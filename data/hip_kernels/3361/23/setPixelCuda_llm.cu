#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setPixelCuda( uint8_t * in, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * pointX, uint32_t * pointY, uint32_t pointSize, uint32_t value )
{
    // Use shared memory for point data
    __shared__ uint32_t sharedPointX[256];
    __shared__ uint32_t sharedPointY[256];

    const uint32_t idPoint = blockIdx.x * blockDim.x + threadIdx.x;

    // Load points into shared memory
    if (threadIdx.x < pointSize) {
        sharedPointX[threadIdx.x] = pointX[idPoint];
        sharedPointY[threadIdx.x] = pointY[idPoint];
    }
    __syncthreads();

    if (idPoint < pointSize) {
        const uint32_t x = sharedPointX[threadIdx.x];
        const uint32_t y = sharedPointY[threadIdx.x];
        // Check if point is within bounds and set the pixel
        if (x < width && y < height) {
            in[y * rowSize + x] = value;
        }
    }
}