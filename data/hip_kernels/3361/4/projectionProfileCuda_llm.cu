#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void projectionProfileCuda(const uint8_t *image, uint32_t rowSize, bool horizontal, uint32_t width, uint32_t height, uint32_t *projection)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Changed if condition to guard thread execution using thread index check
    if (x < width && y < height) {
        uint32_t index = y * rowSize + x;
        atomicAdd(&projection[image[index]], 1); // Use atomic operations for synchronization
    }
}