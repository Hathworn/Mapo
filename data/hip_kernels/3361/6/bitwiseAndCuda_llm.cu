#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseAndCuda(const uint8_t* __restrict__ in1, uint32_t rowSizeIn1, const uint8_t* __restrict__ in2, uint32_t rowSizeIn2, uint8_t* __restrict__ out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate unique thread coordinates
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure thread is within image bounds
    if (x < width && y < height) {
        // Precompute index variables for simplicity
        const uint32_t idIn1 = y * rowSizeIn1 + x;
        const uint32_t idIn2 = y * rowSizeIn2 + x;
        const uint32_t idOut = y * rowSizeOut + x;

        // Perform bitwise AND operation
        out[idOut] = in1[idIn1] & in2[idIn2];
    }
}