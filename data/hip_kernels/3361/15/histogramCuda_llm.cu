#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogramCuda(const uint8_t *data, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t *histogram) {
    // Calculate unique thread index within the image area
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within image bounds
    if (x < width && y < height) {
        // Compute linear index to access the data
        const uint32_t id = y * rowSize + x;

        // Use atomic operation to safely update the histogram
        atomicAdd(&histogram[data[id]], 1);
    }
}