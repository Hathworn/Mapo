#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void maximumCuda(const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global thread index in both dimensions
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure the thread is within image bounds
    if (x < width && y < height) {
        const uint32_t idx = y * rowSizeOut + x; // Flatten index calculation for improved access
        out[idx] = max(in1[y * rowSizeIn1 + x], in2[y * rowSizeIn2 + x]); // Use built-in max function for clarity
    }
}