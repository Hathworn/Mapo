#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtractCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate the 1D index for the pixel
    const uint32_t index = blockDim.x * blockIdx.x + threadIdx.x + 
                           (blockDim.y * blockIdx.y + threadIdx.y) * rowSizeOut;

    // Check if the index is in bounds
    if (index < width * height) {
        // Load input pixels
        const uint8_t in1Val = in1[index];
        const uint8_t in2Val = in2[index];

        // Perform subtraction and write the result
        out[index] = (in1Val > in2Val) ? (in1Val - in2Val) : 0;
    }
}