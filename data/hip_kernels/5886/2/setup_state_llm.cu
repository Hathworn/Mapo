#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique thread index
    hiprand_init(seed, idx, 0, &state[idx]); // Initialize state with unique index
}