#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// libs

__global__ void render_init(int mx, int my, hiprandState *randState, int seed) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure all threads within the block initialize their state
    if (i < mx && j < my) {
        int pixel_index = j * mx + i;
        // Different seed for each thread using pixel_index
        hiprand_init(seed + pixel_index, pixel_index, 0, &randState[pixel_index]);
    }
}