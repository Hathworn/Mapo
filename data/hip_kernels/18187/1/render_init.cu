#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// libs



__global__ void render_init(int mx, int my, hiprandState *randState, int seed) {
if (threadIdx.x == 0 && threadIdx.y == 0) {
hiprand_init(seed, 0, 0, randState);
}
int i = threadIdx.x + blockIdx.x * blockDim.x;
int j = threadIdx.y + blockIdx.y * blockDim.y;

if ((i >= mx) || (j >= my)) {
return;
}
int pixel_index = j * mx + i;
// same seed, different index
hiprand_init(seed + pixel_index, pixel_index, 0,
&randState[pixel_index]);
}