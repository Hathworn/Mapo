#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addVect(int *vect1, int *vect2, int *resultVect) {
    // Use local thread index with grid-stride loop for better access pattern and work distribution
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < N; i += blockDim.x * gridDim.x) {
        resultVect[i] = vect1[i] + vect2[i];
    }
}