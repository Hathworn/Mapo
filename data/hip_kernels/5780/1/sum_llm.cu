#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum(int* input, int* sumOut) {
    int blockOffset = blockIdx.x * blockDim.x;
    int globalIdx = blockOffset + threadIdx.x;

    for (int j = 0; j < 100 / (blockDim.x * gridDim.x); j++) {
        int idx = globalIdx + (j * blockDim.x * gridDim.x);
        if (idx < 100) {
            atomicAdd(sumOut, input[idx]);  // Consolidate index calculations
            printf("NUM:%d Thread: %d ||\n", input[idx], globalIdx);  // Simplify printf index
        }
    }
}