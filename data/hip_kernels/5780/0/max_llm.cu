#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void max(int* input, int* maxOut) {
    // Calculate global thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop unrolling for improved performance
    for(int offset = idx; offset < 100; offset += stride){
        atomicMax(maxOut, input[offset]);
        printf("NUM:%d Thread: %d ||\n", input[offset], idx);
    }
}