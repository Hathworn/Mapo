#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate thread index once
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total threads only once and store
    const int numThreads = blockDim.x * gridDim.x;

    // Iterate over output indices, skip computed tiles for efficiency
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Directly index into src and tgt
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}