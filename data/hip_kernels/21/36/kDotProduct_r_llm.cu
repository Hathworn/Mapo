#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define WARP_SIZE 32

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Loop unrolling for better performance
    if (eidx < numCols) {
        for (; eidx < numElements; eidx += numCols) {
            sum += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Reduce using sequential addressing, avoiding shared memory bank conflicts
    for (uint stride = DP_BLOCKSIZE / 2; stride >= WARP_SIZE; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Warp-level reduction without __syncthreads() for better performance
    if (threadIdx.x < WARP_SIZE) {
        volatile float* mysh = shmem;
        #pragma unroll
        for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1) {
            mysh[threadIdx.x] += mysh[threadIdx.x + offset];
        }
        // Writing final result to target
        if (threadIdx.x == 0) {
            target[blockIdx.x] = mysh[0];
        }
    }
}