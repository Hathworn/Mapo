#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Use block-level parallelism for better performance
    const uint tidx = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;
    /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(seed, tidx, 0, &state[tidx]);
}