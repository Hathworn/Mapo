#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * blockDim.x + threadIdx.x;  // Use blockDim.x for generalization

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        if (labelp != maxp) {
            correctProbs[tx] = 0.0f;
        } else {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp);  // Use explicit parentheses
            }
            correctProbs[tx] = 1.0f / static_cast<float>(numMax);  // Use static_cast for clarity
        }
    }
}