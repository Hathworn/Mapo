#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_matmul(float* A, float* B, float* C, size_t n)
{
    float sum = 0.0f;

#ifndef MATMUL_USE_SHARED
    int ia = (blockDim.y * blockIdx.y + threadIdx.y) * n;
    int ib = blockDim.x * blockIdx.x + threadIdx.x;
    int ic = ia + ib;

    // Multiply two matrices
    for (int k = 0; k < n; k += BLOCK_SIZE) {
        sum += A[ia + k] * B[ib + k * n];
    }
#else
    int ia = (blockDim.y * blockIdx.y) * n;
    int ib = blockDim.x * blockIdx.x;
    int tileidx = n * threadIdx.y + threadIdx.x;
    int ic = ia + ib + tileidx;

    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Iterate over tiles efficiently
    for (int tileBase = 0; tileBase < n; tileBase += BLOCK_SIZE) {
        // Load tiles into shared memory
        As[threadIdx.y][threadIdx.x] = A[ia + tileBase + tileidx];
        Bs[threadIdx.y][threadIdx.x] = B[ib + tileBase * n + tileidx];
        
        __syncthreads(); // Ensure the tiles are loaded before computing
        
        // Perform multiplication on tiles
        for (int k = 0; k < BLOCK_SIZE; k++) {
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }
        
        __syncthreads(); // Ensure computations are finished before next iteration
    }
#endif
    
    C[ic] = sum; // Write result to global memory
}