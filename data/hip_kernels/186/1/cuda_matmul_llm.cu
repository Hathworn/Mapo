#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_matmul(float* A, float* B, float* C, size_t n)
{
    float sum = 0.0f;

#ifndef MATMUL_USE_SHARED
    int ia = (blockDim.y * blockIdx.y + threadIdx.y) * n;
    int ib = blockDim.x * blockIdx.x + threadIdx.x;
    int ic = ia + ib;

    // Multiply two matrices
    for (int k = 0; k < n; k++)
        sum += A[ia + k] * B[ib + k * n];
#else
    int ia = (blockDim.y * blockIdx.y) * n;
    int ib = blockDim.x * blockIdx.x;
    int tileidx = n * threadIdx.y + threadIdx.x;
    int ic = ia + ib + tileidx;

    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    for (int aoff = 0, boff = 0; aoff < n; aoff += BLOCK_SIZE, boff += BLOCK_SIZE * n)
    {
        // Load the "tile" matrices from global memory to shared memory
        As[threadIdx.y][threadIdx.x] = (aoff + threadIdx.x < n) ? A[ia + aoff + tileidx] : 0.0f;
        Bs[threadIdx.y][threadIdx.x] = (boff + threadIdx.y * n < n * n) ? B[ib + boff + tileidx] : 0.0f;

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices
        for (int k = 0; k < BLOCK_SIZE; k++)
            sum += As[threadIdx.y][k] * Bs[k][threadIdx.x];

        // Synchronize to make sure that the preceding computation is done
        __syncthreads();
    }
#endif
    C[ic] = sum; // Write the result
}