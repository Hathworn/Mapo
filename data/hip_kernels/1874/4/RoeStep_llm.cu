#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void d_boundaryCondition(const int nbrOfGrids, double *d_u1, double *d_u2, double *d_u3) {
    d_u1[0] = d_u1[1];
    d_u2[0] = -d_u2[1];
    d_u3[0] = d_u3[1];
    d_u1[nbrOfGrids - 1] = d_u1[nbrOfGrids - 2];
    d_u2[nbrOfGrids - 1] = -d_u2[nbrOfGrids - 2];
    d_u3[nbrOfGrids - 1] = d_u3[nbrOfGrids - 2];
}

__global__ void RoeStep(const int nbrOfGrids, double *d_u1, double *d_u2, double *d_u3, const double *d_vol, double *d_f1, double *d_f2, double *d_f3, const double *d_tau, const double *d_h, const double *d_gama, double *w1, double *w2, double *w3, double *w4, double *fc1, double *fc2, double *fc3, double *fr1, double *fr2, double *fr3, double *fl1, double *fl2, double *fl3, double *fludif1, double *fludif2, double *fludif3, double *rsumr, double *utilde, double *htilde, double *uvdif, double *absvt, double *ssc, double *vsc, double *eiglam1, double *eiglam2, double *eiglam3, double *sgn1, double *sgn2, double *sgn3, int *isb1, int *isb2, int *isb3, double *a1, double *a2, double *a3, double *ac11, double *ac12, double *ac13, double *ac21, double *ac22, double *ac23) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Grid stride loop for better performance

    for (int i = index; i < nbrOfGrids; i += stride) {
        // Compute parameter vector w
        w1[i] = sqrt(d_vol[i] * d_u1[i]);
        w2[i] = w1[i] * d_u2[i] / d_u1[i];
        w4[i] = (*d_gama - 1) * (d_u3[i] - 0.5 * d_u2[i] * d_u2[i] / d_u1[i]);
        w3[i] = w1[i] * (d_u3[i] + w4[i]) / d_u1[i];

        // Compute center fluxes
        fc1[i] = w1[i] * w2[i];
        fc2[i] = w2[i] * w2[i] + d_vol[i] * w4[i];
        fc3[i] = w2[i] * w3[i];
    }

    __syncthreads(); // Sync threads after computing center fluxes

    for (int i = index; i < nbrOfGrids; i += stride) {
        if (i > 0) {
            // Compute wall fluxes
            fl1[i] = fc1[i - 1]; fr1[i] = fc1[i];
            fl2[i] = fc2[i - 1]; fr2[i] = fc2[i];
            fl3[i] = fc3[i - 1]; fr3[i] = fc3[i];

            // Compute flux differences
            fludif1[i] = fr1[i] - fl1[i];
            fludif2[i] = fr2[i] - fl2[i];
            fludif3[i] = fr3[i] - fl3[i];
        }
    }

    __syncthreads(); // Sync before computing state variables

    for (int i = index; i < nbrOfGrids; i += stride) {
        if (i > 0) {
            // Compute mean interface state variables
            rsumr[i] = 1 / (w1[i - 1] + w1[i]);
            utilde[i] = (w2[i - 1] + w2[i]) * rsumr[i];
            htilde[i] = (w3[i - 1] + w3[i]) * rsumr[i];
            absvt[i] = 0.5 * utilde[i] * utilde[i];
            uvdif[i] = utilde[i] * fludif2[i];

            // Compute speed of sound
            ssc[i] = (*d_gama - 1) * (htilde[i] - absvt[i]);
            vsc[i] = ssc[i] > 0.0 ? sqrt(ssc[i]) : sqrt(abs(ssc[i]));
        }
    }

    __syncthreads(); // Sync before eigenvalues and projections

    for (int i = index; i < nbrOfGrids; i += stride) {
        if (i > 0) {
            // Compute eigenvalues and projections
            eiglam1[i] = utilde[i] - vsc[i];
            eiglam2[i] = utilde[i];
            eiglam3[i] = utilde[i] + vsc[i];
            sgn1[i] = eiglam1[i] < 0.0 ? -1 : 1;
            sgn2[i] = eiglam2[i] < 0.0 ? -1 : 1;
            sgn3[i] = eiglam3[i] < 0.0 ? -1 : 1;
            a1[i] = 0.5 * ((*d_gama - 1) * (absvt[i] * fludif1[i] + fludif3[i] - uvdif[i]) - vsc[i] * (fludif2[i] - utilde[i] * fludif1[i])) / ssc[i];
            a2[i] = (*d_gama - 1) * ((htilde[i] - 2 * absvt[i]) * fludif1[i] + uvdif[i] - fludif3[i]) / ssc[i];
            a3[i] = 0.5 * ((*d_gama - 1) * (absvt[i] * fludif1[i] + fludif3[i] - uvdif[i]) + vsc[i] * (fludif2[i] - utilde[i] * fludif1[i])) / ssc[i];
        }
    }

    __syncthreads(); // Sync before adjusting projection coefficients

    for (int i = index; i < nbrOfGrids; i += stride) {
        if (i > 0) {
            // Adjust projection coefficients
            a1[i] /= eiglam1[i] + tiny;
            a2[i] /= eiglam2[i] + tiny;
            a3[i] /= eiglam3[i] + tiny;

            ac11[i] = -sgn1[i] * a1[i] * eiglam1[i];
            ac12[i] = -sgn2[i] * a2[i] * eiglam2[i];
            ac13[i] = -sgn3[i] * a3[i] * eiglam3[i];

            double dtdx = *d_tau / *d_h;
            if ((i > 1) && (i < nbrOfGrids - 1)) {
                isb1[i] = i - int(sgn1[i]);
                ac21[i] = ac11[i] + eiglam1[i] * ((fmax(0.0, fmin(sbpar1 * a1[isb1[i]], fmax(a1[i], fmin(a1[isb1[i]], sbpar2 * a1[i])))) + fmin(0.0, fmax(sbpar1 * a1[isb1[i]], fmin(a1[i], fmax(a1[isb1[i]], sbpar2 * a1[i]))))) * (sgn1[i] - dtdx * eiglam1[i]));
                isb2[i] = i - int(sgn2[i]);
                ac22[i] = ac12[i] + eiglam2[i] * ((fmax(0.0, fmin(sbpar1 * a2[isb2[i]], fmax(a2[i], fmin(a2[isb2[i]], sbpar2 * a2[i])))) + fmin(0.0, fmax(sbpar1 * a2[isb2[i]], fmin(a2[i], fmax(a2[isb2[i]], sbpar2 * a2[i]))))) * (sgn2[i] - dtdx * eiglam2[i]));
                isb3[i] = i - int(sgn3[i]);
                ac23[i] = ac13[i] + eiglam3[i] * ((fmax(0.0, fmin(sbpar1 * a3[isb3[i]], fmax(a3[i], fmin(a3[isb3[i]], sbpar2 * a3[i])))) + fmin(0.0, fmax(sbpar1 * a3[isb3[i]], fmin(a3[i], fmax(a3[isb3[i]], sbpar2 * a3[i]))))) * (sgn3[i] - dtdx * eiglam3[i]));
            }
        }
    }

    __syncthreads(); // Sync before computing final fluxes

    for (int i = index; i < nbrOfGrids; i += stride) {
        if (i > 0) {
            // Compute final fluxes
            d_f1[i] = 0.5 * (fl1[i] + fr1[i] + ac21[i] + ac22[i] + ac23[i]);
            d_f2[i] = 0.5 * (fl2[i] + fr2[i] + eiglam1[i] * ac21[i] + eiglam2[i] * ac22[i] + eiglam3[i] * ac23[i]);
            d_f3[i] = 0.5 * (fl3[i] + fr3[i] + (htilde[i] - utilde[i] * vsc[i]) * ac21[i] + absvt[i] * ac22[i] + (htilde[i] + utilde[i] * vsc[i]) * ac23[i]);

            if (i < nbrOfGrids - 1) {
                // Update variables U
                d_u1[i] -= *d_tau / *d_h * (d_f1[i + 1] - d_f1[i]);
                d_u2[i] -= *d_tau / *d_h * (d_f2[i + 1] - d_f2[i]);
                d_u3[i] -= *d_tau / *d_h * (d_f3[i + 1] - d_f3[i]);
            }
        }
    }

    __syncthreads(); // Sync before applying boundary conditions if needed

    if (index == 0) {
        d_boundaryCondition(nbrOfGrids, d_u1, d_u2, d_u3);
    }
}