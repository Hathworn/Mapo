#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void boundaryCondition(const int nbrOfGrids, double *d_u1, double *d_u2, double *d_u3) {
    // Use shared memory for improved memory access efficiency
    extern __shared__ double sharedMem[];
    double *s_u1 = sharedMem;
    double *s_u2 = &sharedMem[nbrOfGrids];
    double *s_u3 = &sharedMem[nbrOfGrids * 2];

    int threadId = threadIdx.x;

    // Boundary conditions: only a single thread is needed for these updates
    if (threadId == 0) {
        // Copy data from global to shared memory
        s_u1[0] = d_u1[1];
        s_u2[0] = -d_u2[1];
        s_u3[0] = d_u3[1];

        // Apply boundary conditions on the other end
        s_u1[nbrOfGrids - 1] = d_u1[nbrOfGrids - 2];
        s_u2[nbrOfGrids - 1] = -d_u2[nbrOfGrids - 2];
        s_u3[nbrOfGrids - 1] = d_u3[nbrOfGrids - 2];

        // Copy data back to global memory
        d_u1[0] = s_u1[0];
        d_u2[0] = s_u2[0];
        d_u3[0] = s_u3[0];
        d_u1[nbrOfGrids - 1] = s_u1[nbrOfGrids - 1];
        d_u2[nbrOfGrids - 1] = s_u2[nbrOfGrids - 1];
        d_u3[nbrOfGrids - 1] = s_u3[nbrOfGrids - 1];
    }
}