#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void updateCMax(const int nbrOfGrids, const double *d_u1, const double *d_u2, const double *d_u3, const double *d_gama, double *d_cMax) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    double ro, p, u;
    __shared__ double s_cMax;  // Use shared memory to store max result for the block
    s_cMax = 0;  // Initialize shared max value

    for (int i = index; i < nbrOfGrids; i += stride) {
        if (d_u1[i] == 0) continue;
        ro = d_u1[i];
        u = d_u2[i] / ro;
        p = (d_u3[i] - ro * u * u / 2) * (*d_gama - 1);
        double c = sqrt(*d_gama * abs(p) / ro);
        double localMax = c + abs(u);
        atomicMax((double*)&s_cMax, localMax);  // Use atomic operation for max
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        atomicMax(d_cMax, s_cMax);  // Update global max from shared max
    }
}

__global__ void updateTau(const int nbrOfGrids, const double *d_u1, const double *d_u2, const double *d_u3, const double *d_gama, double *d_cMax, const double *d_h, const double *d_cfl, double *d_tau) {
    updateCMax(nbrOfGrids, d_u1, d_u2, d_u3, d_gama, d_cMax);
    *d_tau = *d_cfl * *d_h / *d_cMax;
}