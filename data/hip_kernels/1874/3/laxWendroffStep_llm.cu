#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void updateU(const int nbrOfGrids, double *d_u1, double *d_u2, double *d_u3, const double *d_u1Temp, const double *d_u2Temp, const double *d_u3Temp) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Use gridDim for correct stride
    for (int i = index; i < nbrOfGrids; i += stride) {
        if ((i > 0) && (i < nbrOfGrids - 1)) {
            d_u1[i] = d_u1Temp[i];
            d_u2[i] = d_u2Temp[i];
            d_u3[i] = d_u3Temp[i];
        }
    }
}

__device__ void step(const int nbrOfGrids, const double *d_u1, const double *d_u2, const double *d_u3, double *d_u1Temp, double *d_u2Temp, double *d_u3Temp, const double *d_f1, const double *d_f2, const double *d_f3, const double *d_tau, const double *d_h) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Use gridDim for correct stride
    double tau_h = *d_tau / *d_h;
    for (int i = index; i < nbrOfGrids; i += stride) {
        if ((i > 0) && (i < nbrOfGrids - 1)) {
            d_u1Temp[i] = d_u1[i] - tau_h * (d_f1[i] - d_f1[i - 1]);
            d_u2Temp[i] = d_u2[i] - tau_h * (d_f2[i] - d_f2[i - 1]);
            d_u3Temp[i] = d_u3[i] - tau_h * (d_f3[i] - d_f3[i - 1]);
        }
    }
}

__device__ void halfStep(const int nbrOfGrids, const double *d_u1, const double *d_u2, const double *d_u3, double *d_u1Temp, double *d_u2Temp, double *d_u3Temp, const double *d_f1, const double *d_f2, const double *d_f3, const double *d_tau, const double *d_h) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Use gridDim for correct stride
    double tau_h_half = *d_tau / (2.0 * *d_h); 
    for (int i = index; i < nbrOfGrids; i += stride) {
        if ((i > 0) && (i < nbrOfGrids - 1)) {
            d_u1Temp[i] = (d_u1[i + 1] + d_u1[i]) / 2 - tau_h_half * (d_f1[i + 1] - d_f1[i]);
            d_u2Temp[i] = (d_u2[i + 1] + d_u2[i]) / 2 - tau_h_half * (d_f2[i + 1] - d_f2[i]);
            d_u3Temp[i] = (d_u3[i + 1] + d_u3[i]) / 2 - tau_h_half * (d_f3[i + 1] - d_f3[i]);
        }
    }
}

__device__ void updateFlux(const int nbrOfGrids, const double *d_u1, const double *d_u2, const double *d_u3, double *d_f1, double *d_f2, double *d_f3, const double *d_gama) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Use gridDim for correct stride
    double gamma_minus_one = *d_gama - 1.0;
    for (int i = index; i < nbrOfGrids; i += stride) {
        double rho = d_u1[i];
        double m = d_u2[i];
        double e = d_u3[i];
        double p = gamma_minus_one * (e - m * m / (2 * rho)); // Avoid repeated division
        d_f1[i] = m;
        d_f2[i] = m * m / rho + p;
        d_f3[i] = m / rho * (e + p);
    }
}

__device__ void d_boundaryCondition(const int nbrOfGrids, double *d_u1, double *d_u2, double *d_u3) {
    d_u1[0] = d_u1[1];
    d_u2[0] = -d_u2[1];
    d_u3[0] = d_u3[1];
    d_u1[nbrOfGrids - 1] = d_u1[nbrOfGrids - 2];
    d_u2[nbrOfGrids - 1] = -d_u2[nbrOfGrids - 2];
    d_u3[nbrOfGrids - 1] = d_u3[nbrOfGrids - 2];
}

__global__ void laxWendroffStep(const int nbrOfGrids, double *d_u1, double *d_u2, double *d_u3, double *d_u1Temp, double *d_u2Temp, double *d_u3Temp, double *d_f1, double *d_f2, double *d_f3, const double *d_tau, const double *d_h, const double *d_gama) {
    updateFlux(nbrOfGrids, d_u1, d_u2, d_u3, d_f1, d_f2, d_f3, d_gama);
    halfStep(nbrOfGrids, d_u1, d_u2, d_u3, d_u1Temp, d_u2Temp, d_u3Temp, d_f1, d_f2, d_f3, d_tau, d_h);
    d_boundaryCondition(nbrOfGrids, d_u1Temp, d_u2Temp, d_u3Temp);
    updateFlux(nbrOfGrids, d_u1Temp, d_u2Temp, d_u3Temp, d_f1, d_f2, d_f3, d_gama);
    step(nbrOfGrids, d_u1, d_u2, d_u3, d_u1Temp, d_u2Temp, d_u3Temp, d_f1, d_f2, d_f3, d_tau, d_h);
    updateU(nbrOfGrids, d_u1, d_u2, d_u3, d_u1Temp, d_u2Temp, d_u3Temp);
    d_boundaryCondition(nbrOfGrids, d_u1, d_u2, d_u3);
}