#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void updateCMax(const int nbrOfGrids, const double *d_u1, const double *d_u2, const double *d_u3, const double *d_gama, double *d_cMax)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    __shared__ double sharedMax[256]; // Use shared memory for partial max values
    double localMax = 0; // Local max for this thread
    double ro, p, u;
    
    for (int i = index; i < nbrOfGrids; i += stride) {
        if (d_u1[i] == 0) continue;
        ro = d_u1[i];
        u = d_u2[i] / ro;
        p = (d_u3[i] - ro * u * u / 2) * (*d_gama - 1);
        double c = sqrt(*d_gama * abs(p) / ro);
        double speed = c + abs(u);
        if (localMax < speed) localMax = speed; // Update local max
    }
    
    sharedMax[threadIdx.x] = localMax; // Each thread writes its local max to shared memory
    __syncthreads(); // Ensure all threads have finished writing to shared
    
    // Perform reduction to find the maximum speed in the block
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sharedMax[threadIdx.x] = fmax(sharedMax[threadIdx.x], sharedMax[threadIdx.x + s]);
        }
        __syncthreads();
    }
    
    if (threadIdx.x == 0) atomicMax(d_cMax, sharedMax[0]); // Use atomicMax to update global max
}

__global__ void initDeviceMemory(const int nbrOfGrids, double *d_u1, double *d_u2, double *d_u3, double *d_vol, double *d_h, double *d_length, double *d_gama, double *d_cfl, double *d_nu, double *d_tau, double *d_cMax, double *d_t) {
    *d_t = 0; // time
    *d_length = 1; // length of shock tube
    *d_gama = 1.4; // ratio of specific heats
    *d_cfl = 0.9; // Courant-Friedrichs-Lewy number
    *d_nu = 0.0; // artificial viscosity coefficient
    *d_h = *d_length / (nbrOfGrids - 1); // space grid size
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x;
    
    for (int i = index; i < nbrOfGrids; i += stride) {
        double e, ro, p, u = 0;
        if (i < nbrOfGrids) {
            if (i >= int(nbrOfGrids / 2)) { ro = 0.125, p = 0.1; }
            else { ro = 1, p = 1; }
            e = p / (*d_gama - 1) + ro * u * u / 2;
            d_u1[i] = ro;
            d_u2[i] = ro * u;
            d_u3[i] = e;
            d_vol[i] = 1;
        }
    }
    
    updateCMax(nbrOfGrids, d_u1, d_u2, d_u3, d_gama, d_cMax);
    *d_tau = (*d_cfl) * (*d_h) / (*d_cMax); // initial time grid size, It will be modified to tMax if this > tMax
}