#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpuIt3(float *tNew, float *tOld, float *tOrig, int x, int y, int z, float k, float st) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < x*y*z) {
        float center = tOld[i];
        float front = (i >= x*y) ? tOld[i - x*y] : center;
        float back = (i < x*y*(z-1)) ? tOld[i + x*y] : center;
        float top = (i >= x) ? tOld[i - x] : center;
        float bottom = (i < x*y*z - x) ? tOld[i + x] : center;
        float left = (i % x > 0) ? tOld[i - 1] : center;
        float right = (i % x < x-1) ? tOld[i + 1] : center;

        // Compute the new temperature value
        tNew[i] = center + k * (front + back + top + bottom + left + right - 6 * center);

        // Replace with original stability value if not a heat source
        if (tOrig[i] != st) {
            tNew[i] = tOrig[i];
        }
    }
}