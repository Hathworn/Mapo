#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpuIt(float *tNew,float *tOld,float *tOrig,int x,int y,int z,float k,float st) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // Boundary check to avoid accessing out-of-bounds memory
    if(i < x*y*z){
        int xi = i % x; // X index
        int yi = (i / x) % y; // Y index

        // Neighbors' indices with boundary conditions
        int left   = (xi > 0)    ? i - 1   : i; 
        int right  = (xi < x-1)  ? i + 1   : i;
        int top    = (yi > 0)    ? i - x   : i;
        int bottom = (yi < y-1)  ? i + x   : i;

        // Compute update for interior and boundary cells
        tNew[i] = tOld[i] + k * (tOld[left] + tOld[right] + tOld[top] + tOld[bottom] - 4 * tOld[i]);

        // Restore original temperature if not a heat source
        if(tOrig[i] != st){
            tNew[i] = tOrig[i];
        }
    }
}