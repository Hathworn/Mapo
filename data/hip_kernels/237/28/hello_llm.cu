#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello() {
    // Use a single thread to print message to avoid concurrency issues
    if (threadIdx.x == 0) {
        printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
    }
}