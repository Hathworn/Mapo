#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf( unsigned int* d_input_cdf, float* d_output_cdf, int n )
{
    // Fetch once to reduce global memory access
    const float normalization_constant = 1.f / d_input_cdf[n - 1];
    
    int global_index_1d = ( blockIdx.x * blockDim.x ) + threadIdx.x;

    if ( global_index_1d < n )
    {
        // Load input value once and use
        float output_value = d_input_cdf[global_index_1d] * normalization_constant;
        d_output_cdf[global_index_1d] = output_value;
    }
}