#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tonemap(float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new, float min_log_Y, float max_log_Y, float log_Y_range, int num_bins, int num_pixels_y, int num_pixels_x)
{
    int nx = num_pixels_x;
    int ny = num_pixels_y;
    int2 image_index_2d = make_int2((blockIdx.x * blockDim.x) + threadIdx.x, (blockIdx.y * blockDim.y) + threadIdx.y);
    int image_index_1d = (nx * image_index_2d.y) + image_index_2d.x;

    // Ensure thread is within image bounds
    if (image_index_2d.x < nx && image_index_2d.y < ny)
    {
        float x = d_x[image_index_1d];
        float y = d_y[image_index_1d];
        float log_Y = d_log_Y[image_index_1d];
        int bin_index = min(num_bins - 1, static_cast<int>((num_bins * (log_Y - min_log_Y)) / log_Y_range));
        float Y_new = d_cdf_norm[bin_index];

        // Perform calculations once; reuse to save computations
        float Y_div_y = Y_new / y;
        float X_new = x * Y_div_y;
        float Z_new = (1 - x - y) * Y_div_y;

        // Use floats instead of multiple constants
        float r_new = fmaf(X_new, 3.2406f, fmaf(Y_new, -1.5372f, Z_new * -0.4986f));
        float g_new = fmaf(X_new, -0.9689f, fmaf(Y_new, 1.8758f, Z_new * 0.0415f));
        float b_new = fmaf(X_new, 0.0557f, fmaf(Y_new, -0.2040f, Z_new * 1.0570f));

        d_r_new[image_index_1d] = r_new;
        d_g_new[image_index_1d] = g_new;
        d_b_new[image_index_1d] = b_new;
    }
}