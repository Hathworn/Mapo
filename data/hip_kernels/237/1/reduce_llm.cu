#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int* s) {
    // Use warp shuffle operations for efficiency
    p += __shfl_down_sync(0xFFFFFFFF, p, 16);
    p += __shfl_down_sync(0xFFFFFFFF, p, 8);
    p += __shfl_down_sync(0xFFFFFFFF, p, 4);
    p += __shfl_down_sync(0xFFFFFFFF, p, 2);
    p += __shfl_down_sync(0xFFFFFFFF, p, 1);
    
    s[threadIdx.x] = p;  // Store reduction result in shared memory
    return s[0];         // Result is in s[0] due to shuffle reduction
}

__global__ void reduce(unsigned int* d_out_shared, const unsigned int* d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    if (t == 0) {
        *d_out_shared = sr;
    }
}