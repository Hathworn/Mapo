#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth(float * v_new, const float * v) {
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;  // Fix indexing to correct order
    int numThreads = blockDim.x * gridDim.x;
    
    if (myIdx < numThreads) {  // Bounds checking for safety
        int myLeftIdx = max(myIdx - 1, 0);  // Use max for left boundary
        int myRightIdx = min(myIdx + 1, numThreads - 1);  // Use min for right boundary
        float myElt = v[myIdx];
        float myLeftElt = v[myLeftIdx];
        float myRightElt = v[myRightIdx];
        v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
    }
}