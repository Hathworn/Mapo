#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // You are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid  = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    // Optimization: Unroll loops and fine-tune shared memory access
    for (int stage = 0; stage <= 5; ++stage)
    {
        int stride = 1 << stage;
        int pairDistance = stride >> 1;
        bool ascending = ((tid & stride == 0));

        for (int substage = stage; substage >= 0; --substage)
        {
            int pos = 2 * tid - (tid & pairDistance);
            if (pos + pairDistance < 64)
            {
                float valA = sdata[pos];
                float valB = sdata[pos + pairDistance];
                if ((valA > valB) == ascending)
                {
                    sdata[pos] = valB;
                    sdata[pos + pairDistance] = valA;
                }
            }
            __syncthreads();
            pairDistance >>= 1;
        }
    }

    d_out[tid] = sdata[tid];
}