#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Calculate global thread ID
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Using global thread ID for output
    printf("Hello world! I'm thread %d\n", global_id);
}