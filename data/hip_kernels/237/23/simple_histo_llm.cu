#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;

    // Check if myId is within bounds to avoid out of bounds access
    if (myId < BIN_COUNT) {
        int myItem = d_in[myId];
        int myBin = myItem % BIN_COUNT;

        // Use atomic add to prevent race conditions
        atomicAdd(&(d_bins[myBin]), 1);
    }
}