#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

typedef unsigned char byte;

unsigned greyScale(char* str, byte*& image, byte*& pixels, unsigned int& width, unsigned int& height);
unsigned writeImage(byte*& image, unsigned int& width, unsigned int& height, char* str);
void sobel(byte*& image, byte*& edged, unsigned int& width, unsigned int& height);
void sobelOpenMP(byte*& image, byte*& edged, unsigned int& width, unsigned int& height);

__global__ void sobelCuda(byte* image, byte* edged, int width, int height) {
    // Calculate the x and y coordinates of the pixel
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int j = x / width;
    int i = x % width;

    // Check boundary conditions
    if (i < 1 || i >= (width - 1) || j < 1 || j >= (height - 1))
        return;

    // Precompute indices for efficiency
    int index = i + j * width;
    int idx1 = index - 1;
    int idx3 = index + 1;
    int idx4 = index - width;
    int idx5 = index + width;
    int idx2 = idx4 - 1;
    int idx6 = idx4 + 1;
    int idx7 = idx5 - 1;
    int idx8 = idx5 + 1;

    // Compute gradient in X direction
    int gX = -image[idx2] - 2 * image[idx4] - image[idx6]
             + image[idx7] + 2 * image[idx5] + image[idx8];

    // Compute gradient in Y direction
    int gY = -image[idx2] + image[idx6]
             - 2 * image[idx1] + 2 * image[idx3]
             - image[idx7] + image[idx8];

    // Compute the edge intensity and clamp to 255
    edged[index] = (byte)min(sqrt((float)(gX * gX) + (gY * gY)), 255.0f);
}