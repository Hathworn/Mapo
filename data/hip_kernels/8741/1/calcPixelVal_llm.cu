#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use __restrict__ to suggest to the compiler that pointers do not alias
__global__ void calcPixelVal(float* __restrict__ g_idata, const float* __restrict__ constant, const float* __restrict__ min)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a register to store the minimum value to reduce memory access
    float minVal = *min;

    // Optimize boundary check and calculation by merging conditions
    if(i < LENA_SIZE)
    {
        g_idata[i] = (g_idata[i] - minVal) * (*constant);
    }
}