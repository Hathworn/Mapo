#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cube_select(int b, int n, float radius, const float* xyz, int* idx_out) {
    int batch_idx = blockIdx.x;
    xyz += batch_idx * n * 3;
    idx_out += batch_idx * n * 8;
    float judge_dist = radius * radius;

    for(int i = threadIdx.x; i < n; i += blockDim.x) {
        float x = xyz[i * 3];
        float y = xyz[i * 3 + 1];
        float z = xyz[i * 3 + 2];
        
        float temp_dist[8]; // Store smallest distances for each region
        #pragma unroll // Enable loop unrolling for optimization
        for(int j = 0; j < 8; j++) {
            temp_dist[j] = 1e8;
            idx_out[i * 8 + j] = i; // initialize with self index
        }

        for(int j = 0; j < n; j++) {
            if(i == j) continue;

            float tx = xyz[j * 3];
            float ty = xyz[j * 3 + 1];
            float tz = xyz[j * 3 + 2];
            float dist = (x - tx) * (x - tx) + (y - ty) * (y - ty) + (z - tz) * (z - tz);

            if(dist > judge_dist) continue;

            int temp_idx = ((tx > x) << 2) | ((ty > y) << 1) | (tz > z); // Optimize index calculation

            if(dist < temp_dist[temp_idx]) {
                idx_out[i * 8 + temp_idx] = j;
                temp_dist[temp_idx] = dist;
            }
        }
    }
}