#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_index = blockIdx.x;
    idx += m * nsample * batch_index;
    grad_out += m * nsample * c * batch_index;
    grad_points += n * c * batch_index;

    int index = blockIdx.y * blockDim.x + threadIdx.x; // Compute unique thread index across blocks
    int stride = blockDim.x * gridDim.y; // Compute total stride considering multiple blocks

    for (int j = index; j < m * nsample; j += stride) { // Merged loops for better occupancy
        int j_m = j / nsample; // Compute original 'j' in 2D index
        int k = j % nsample;  // Compute original 'k' in 2D index
        int ii = idx[j];
        for (int l = 0; l < c; ++l) {
            atomicAdd(&grad_points[ii * c + l], grad_out[j * c + l]); // Use computed index directly
        }
    }
}