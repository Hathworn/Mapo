#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_index = blockIdx.x;  // Use block index for batch iteration
    int point_index = threadIdx.x; // Use thread index for point iteration

    if (batch_index < b && point_index < m * nsample) {
        int j = point_index / nsample;
        int k = point_index % nsample;
        int ii = idx[j * nsample + k];

        for (int l = 0; l < c; ++l) {
            atomicAdd(&grad_points[ii * c + l], grad_out[j * nsample * c + k * c + l]); // Use atomicAdd for concurrency
        }
    }
}