#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cube_select_four(int b, int n, float radius, const float* xyz, int* idx_out) {
    int batch_idx = blockIdx.x;
    xyz += batch_idx * n * 3;
    idx_out += batch_idx * n * 32;

    float judge_dist = radius * radius;
    __shared__ float shared_xyz[1024 * 3]; // Optimize memory access with shared memory

    for(int i = threadIdx.x; i < n; i += blockDim.x) {
        float x = xyz[i * 3];
        float y = xyz[i * 3 + 1];
        float z = xyz[i * 3 + 2];
        float temp_dist[32];

        // Initialize temporary distances
        for(int j = 0; j < 32; j++) {
            temp_dist[j] = judge_dist;
            idx_out[i * 32 + j] = i; // If not found, just return itself
        }

        // Load xyz data into shared memory
        if (i < 1024) {
            shared_xyz[threadIdx.x * 3] = x;
            shared_xyz[threadIdx.x * 3 + 1] = y;
            shared_xyz[threadIdx.x * 3 + 2] = z;
        }
        __syncthreads();

        for(int j = 0; j < n; j++) {
            if(i == j) continue;
            // Use shared memory for faster access
            float tx = (j < 1024) ? shared_xyz[j * 3] : xyz[j * 3];
            float ty = (j < 1024) ? shared_xyz[j * 3 + 1] : xyz[j * 3 + 1];
            float tz = (j < 1024) ? shared_xyz[j * 3 + 2] : xyz[j * 3 + 2];

            float dist = (x - tx) * (x - tx) + (y - ty) * (y - ty) + (z - tz) * (z - tz);
            if(dist > judge_dist) continue;

            int _x = (tx > x);
            int _y = (ty > y);
            int _z = (tz > z);
            int temp_idx = _x * 16 + _y * 8 + _z * 4;

            bool flag = false;
            for(int k = 0; k < 4; k++) {
                if (dist < temp_dist[temp_idx + k]) {
                    flag = true;
                }
                if (flag) {
                    for (int kk = 3; kk >= k + 1; kk--) {
                        idx_out[i * 32 + temp_idx + kk] = idx_out[i * 32 + temp_idx + kk - 1];
                        temp_dist[temp_idx + kk] = temp_dist[temp_idx + kk - 1];
                    }
                    idx_out[i * 32 + temp_idx + k] = j;
                    temp_dist[temp_idx + k] = dist;
                    break;
                }
            }
        }
    }
}