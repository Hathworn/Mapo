#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    int batch_index = blockIdx.x;
    xyz1 += n * 3 * batch_index;
    xyz2 += m * 3 * batch_index;
    idx += m * nsample * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int j = index; j < m; j += stride) {
        int cnt = 0;
        float x2 = xyz2[j * 3 + 0];
        float y2 = xyz2[j * 3 + 1];
        float z2 = xyz2[j * 3 + 2];
        for (int k = 0; k < n; ++k) {
            if (cnt == nsample)
                break; // only pick the FIRST nsample points in the ball
            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            
            // Use squared distance to avoid sqrt computation
            float dx = x2 - x1;
            float dy = y2 - y1;
            float dz = z2 - z1;
            float d2 = dx * dx + dy * dy + dz * dz; // squared distance

            if (d2 < radius * radius) {
                if (cnt == 0) { // set ALL indices to k, so we have valid indices even if < nsample points are found
                    for (int l = 0; l < nsample; ++l)
                        idx[j * nsample + l] = k;
                }
                idx[j * nsample + cnt] = k;
                cnt += 1;
            }
        }
    }
}