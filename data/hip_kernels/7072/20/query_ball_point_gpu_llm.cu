#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    int batch_idx = blockIdx.x;
    int thread_idx = threadIdx.x;
    int stride = blockDim.x;

    xyz1 += n * 3 * batch_idx;
    xyz2 += m * 3 * batch_idx;
    idx += m * nsample * batch_idx;

    for (int j = thread_idx; j < m; j += stride) {  // Parallel loop over m
        int cnt = 0;
        
        float x2 = xyz2[j * 3 + 0];
        float y2 = xyz2[j * 3 + 1];
        float z2 = xyz2[j * 3 + 2];

        for (int k = 0; k < n; ++k) {
            if (cnt == nsample) break;

            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            float d = sqrtf((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1));

            if (d < radius) {
                if (cnt == 0) {
                    for (int l = 0; l < nsample; ++l)
                        idx[j * nsample + l] = k;
                }
                idx[j * nsample + cnt] = k;
                cnt += 1;
            }
        }
    }
}