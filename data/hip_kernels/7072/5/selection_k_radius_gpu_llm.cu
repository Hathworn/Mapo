#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_k_radius_gpu(int b, int m, int k, float radius, const int* idx, const float* val, int* idx_out, float* val_out){
    int batch_index = blockIdx.x;
    int stride = batch_index * m * k;
    idx += stride;
    val += stride;
    idx_out += stride;
    val_out += stride;

    int i = blockIdx.y * blockDim.x + threadIdx.x;
    if (i < m) {  // Boundary check
        for (int j = 0; j < k; j++) {
            // Simplify condition and unified access for idx and val
            bool within_radius = val[i * k + j] < radius;
            idx_out[i * k + j] = idx[i * k + (within_radius ? j : 0)];
            val_out[i * k + j] = val[i * k + (within_radius ? j : 0)];
        }
    }
}