#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    // Calculate global thread index
    int batch_idx = blockIdx.x;
    int sample_idx = threadIdx.x;

    if (batch_idx < b && sample_idx < nsample * c) {
        int j = sample_idx / c;
        int k = (sample_idx % c);

        for (int i = batch_idx; i < b; i += gridDim.x) {
            if(j < m) {
                int ii = idx[j * nsample + k / c]; // Index calculation based on sample
                int l = k % c;
                out[j * nsample * c + k] = points[ii * c + l];
            }
        }
    }
}