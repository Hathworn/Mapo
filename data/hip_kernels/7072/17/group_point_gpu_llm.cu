#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n * c * batch_index;
    idx += m * nsample * batch_index;
    out += m * nsample * c * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Use shared memory to store points indices for better memory coalescing
    __shared__ int shared_idx[256];
    
    for (int j = index; j < m; j += stride) {
        for (int k = 0; k < nsample; ++k) {
            if (k < 256) {
                shared_idx[k] = idx[j * nsample + k]; // Load indices into shared memory
            }
            __syncthreads();
            
            // Loop through point coordinates
            for (int l = 0; l < c; ++l) {
                int ii = k < 256 ? shared_idx[k] : idx[j * nsample + k];
                out[j * nsample * c + k * c + l] = points[ii * c + l];
            }
        }
    }
}