#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    outi += m * n * batch_index;
    out += m * n * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Copy from dist to dist_out
    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            out[j * n + s] = dist[j * n + s];
            outi[j * n + s] = s;
        }
    }

    float *p_dist;
    // Perform selection sort for the first k elements
    for (int j = index; j < m; j += stride) {
        p_dist = out + j * n;
        for (int s = 0; s < k; ++s) {
            int min = s;

            // Find the min
            for (int t = s + 1; t < n; ++t) {
                if (p_dist[t] < p_dist[min]) {
                    min = t;
                }
            }
            
            // Swap min-th and s-th element
            if (min != s) {
                // Avoid unnecessary memory accesses by using registers
                float tmp = p_dist[min];
                p_dist[min] = p_dist[s];
                p_dist[s] = tmp;
                
                int tmpi = outi[j * n + min];
                outi[j * n + min] = outi[j * n + s];
                outi[j * n + s] = tmpi;
            }
        }
    }
}