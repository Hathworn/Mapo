#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, float *dist, int *idx, float *val) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    idx += m * k * batch_index;
    val += m * k * batch_index;

    // Utilize shared memory for better memory access pattern
    extern __shared__ float shared_dist[];
    
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int j = index; j < m; j += stride) {
        float* p_dist = &shared_dist[j * n];
        // Load distance data to shared memory
        for (int t = threadIdx.x; t < n; t += stride) {
            p_dist[t] = dist[j * n + t];
        }
        __syncthreads();

        // Perform selection sort on shared memory
        for (int s = 0; s < k; ++s) {
            int min = s;
            for (int t = s + 1; t < n; ++t) {
                if (p_dist[t] < p_dist[min]) {
                    min = t;
                }
            }
            idx[j * n + s] = min;
            val[j * n + s] = p_dist[min];
            // Swap min-th and s-th element
            float tmp = p_dist[min];
            p_dist[min] = p_dist[s];
            p_dist[s] = tmp;
        }
        __syncthreads();
    }
}