#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx, int *pts_cnt) {
    int batch_index = blockIdx.x;
    xyz1 += n * 3 * batch_index;
    xyz2 += m * 3 * batch_index;
    idx += m * nsample * batch_index;
    pts_cnt += m * batch_index;

    int index = threadIdx.x; 
    int stride = blockDim.x;

    // Parallelize over each point in the target set
    for (int j = index; j < m; j += stride) {
        int cnt = 0;
        float x2 = xyz2[j * 3 + 0];
        float y2 = xyz2[j * 3 + 1];
        float z2 = xyz2[j * 3 + 2];

        // Unroll the loop to improve memory access efficiency
        #pragma unroll
        for (int k = 0; k < n; ++k) {
            if (cnt == nsample) break;

            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            float dx = x2 - x1;
            float dy = y2 - y1;
            float dz = z2 - z1;
            float d = max(sqrtf(dx * dx + dy * dy + dz * dz), 1e-20f);

            if (d < radius) {
                // Utilize the benefit of filling all indices upfront to avoid branching
                if (cnt == 0) {
                    #pragma unroll
                    for (int l = 0; l < nsample; ++l) {
                        idx[j * nsample + l] = k;
                    }
                }
                idx[j * nsample + cnt] = k;
                cnt += 1;
            }
        }
        pts_cnt[j] = cnt;
    }
}