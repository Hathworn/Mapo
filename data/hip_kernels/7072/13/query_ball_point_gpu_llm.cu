#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    int batch_idx = blockIdx.x;
    if (batch_idx >= b) return; // Check if within batch bounds

    int point_idx = threadIdx.x + blockDim.x * blockIdx.y;
    if (point_idx >= m) return; // Check if within points bounds

    const float x2 = xyz2[point_idx * 3 + 0];
    const float y2 = xyz2[point_idx * 3 + 1];
    const float z2 = xyz2[point_idx * 3 + 2];

    int cnt = 0;
    for (int k = 0; k < n && cnt < nsample; ++k) { // Stop if nsample points are found
        const float x1 = xyz1[k * 3 + 0];
        const float y1 = xyz1[k * 3 + 1];
        const float z1 = xyz1[k * 3 + 2];
        const float d = max(sqrtf((x2-x1)*(x2-x1) + (y2-y1)*(y2-y1) + (z2-z1)*(z2-z1)), 1e-20f);

        if (d < radius) {
            if (cnt == 0) {
                for (int l = 0; l < nsample; ++l)
                    idx[point_idx * nsample + l] = k; // Initialize all indices
            }
            idx[point_idx * nsample + cnt] = k;
            cnt += 1;
        }
    }
}
```
