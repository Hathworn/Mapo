#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;  // Calculate the global thread index

    if (index >= b) return;  // Ensure thread index is within bounds

    // Offset pointers by batch index
    idx += m * nsample * index;
    grad_out += m * nsample * c * index;
    grad_points += n * c * index;

    for (int j = 0; j < m; ++j) {
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            for (int l = 0; l < c; ++l) {
                atomicAdd(&grad_points[ii * c + l], grad_out[j * nsample * c + k * c + l]); // Use atomic operation for safety
            }
        }
    }
}