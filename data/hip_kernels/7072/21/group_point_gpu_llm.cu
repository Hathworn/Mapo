#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    int thread_index = threadIdx.x;
    int batch_stride = n * c;
    int idx_stride = m * nsample;
    int out_stride = m * nsample * c;

    points += batch_stride * batch_index;
    idx += idx_stride * batch_index;
    out += out_stride * batch_index;

    for (int j = thread_index; j < m; j += blockDim.x) {
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            for (int l = 0; l < c; ++l) {
                out[j * nsample * c + k * c + l] = points[ii * c + l];
            }
        }
    }
}