#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        // Use intrinsic function for logarithm to improve performance
        labelLogProbs[tx] = __logf(labelp);

        // Optimize condition check by avoiding unnecessary computations
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            // Use atomic operation to count max-prob labels in parallel
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp);
            }
            // Ensure division is done outside of loop for efficiency
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}