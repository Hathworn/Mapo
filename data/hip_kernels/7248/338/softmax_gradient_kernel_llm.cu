#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel( const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0;

    // Unroll the loop to improve performance
    for (int i = idx; i < dim; i += blockDim.x * 2) {
        tmp += dY[i] * Y[i];
        if(i + blockDim.x < dim) {
            tmp += dY[i + blockDim.x] * Y[i + blockDim.x];
        }
    }
    
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Use tree-based reduction for better performance
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    // Compute gradient
    if (idx == 0) tmp = reduction_buffer[0];
    __syncthreads();
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}