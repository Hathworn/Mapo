#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use a single thread to modify the accuracy value
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
    // Ensure all threads synchronize after division
    __syncthreads();
}