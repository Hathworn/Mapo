#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Use unsigned int for tidx calculation to save some computation
    unsigned int tidx = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialize each thread's state by utilizing faster register storage
    hiprand_init(seed, tidx, 0, &state[tidx]);
}