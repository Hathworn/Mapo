#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    // Use shared memory to reduce repetitive accesses to global memory
    __shared__ float tileCache[1024];
    
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Load a tile into shared memory (assuming the size fits)
        if (threadIdx.x < srcWidth * srcHeight) {
            tileCache[threadIdx.x] = src[threadIdx.x];
        }
        __syncthreads();
        
        // Use the cached value from shared memory
        tgt[i] = tileCache[srcY * srcWidth + srcX];
    }
}