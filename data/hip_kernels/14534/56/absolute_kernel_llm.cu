#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void absolute_kernel(float4* __restrict output, const float4* __restrict input, int elem_count)
{
    int elem_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid stride loop for improved utilization
    for (int i = elem_id; i < elem_count; i += gridDim.x * blockDim.x)
    {
        float4 val = input[i];
        val.x = fabsf(val.x);
        val.y = fabsf(val.y);
        val.z = fabsf(val.z);
        val.w = fabsf(val.w);
        output[i] = val;
    }
}