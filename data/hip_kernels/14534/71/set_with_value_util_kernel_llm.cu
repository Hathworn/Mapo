#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_with_value_util_kernel(double2 * __restrict buf, double v, int elem_count)
{
    int elem_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize: Use memory coalescing and reduce branching within the loop
    if (elem_id >= elem_count) return;

    double2 val = {v, v}; // Initialize using aggregate initialization for better readability
    buf[elem_id] = val;
}