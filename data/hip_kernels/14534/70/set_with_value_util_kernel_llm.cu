#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_with_value_util_kernel(int4 * __restrict buf, int v, int elem_count)
{
    int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
    if (elem_id < elem_count)
    {
        // Use a single constructor call to initialize int4
        buf[elem_id] = make_int4(v, v, v, v);
    }
}