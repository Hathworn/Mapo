#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_with_value_util_kernel(float4 * __restrict buf, float v, int elem_count)
{
    int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
    if (elem_id < elem_count)
    {
        // Precompute the same value to avoid repeated operations
        float4 val = make_float4(v, v, v, v);
        buf[elem_id] = val;
    }
}