#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_buffer_util_kernel(const float4 * __restrict__ input_buf, float4 * __restrict__ output_buf, int elem_count)
{
    // Use shared memory when beneficial
    int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Unroll loop if beneficial
    if (elem_id < elem_count)
    {
        output_buf[elem_id] = input_buf[elem_id];
    }
}