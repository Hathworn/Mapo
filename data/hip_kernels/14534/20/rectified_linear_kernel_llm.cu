#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rectified_linear_kernel(float4* __restrict__ output, const float4* __restrict__ input, float negative_slope, int elem_count)
{
    int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
    if (elem_id < elem_count)
    {
        // Load once to avoid redundant global memory access
        float4 val = input[elem_id];

        // Apply ReLU with negative slope using ternary operator for concise conditional check
        val.x = val.x < 0.0F ? val.x * negative_slope : val.x;
        val.y = val.y < 0.0F ? val.y * negative_slope : val.y;
        val.z = val.z < 0.0F ? val.z * negative_slope : val.z;
        val.w = val.w < 0.0F ? val.w * negative_slope : val.w;

        // Store result back to global memory
        output[elem_id] = val;
    }
}