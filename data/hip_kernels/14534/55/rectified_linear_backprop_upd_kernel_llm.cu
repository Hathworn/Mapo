#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rectified_linear_backprop_upd_kernel( float4 * __restrict input_errors, const float4 * __restrict output_errors, const uint4 * __restrict bits_buffer, float negative_slope, bool add_update_to_destination, int elem_count)
{
    int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
    if (elem_id < elem_count)
    {
        float4 val = output_errors[elem_id];
        uint4 bits = bits_buffer[elem_id >> 5];
        int lane_id = elem_id & 31;
        unsigned int mask = (1 << lane_id);

        // Apply negative slope using early exit to reduce branching
        if ((bits.x & mask) == 0) val.x *= negative_slope;
        if ((bits.y & mask) == 0) val.y *= negative_slope;
        if ((bits.z & mask) == 0) val.z *= negative_slope;
        if ((bits.w & mask) == 0) val.w *= negative_slope;
        
        // Use conditional addition with AddUpdate variable for readability
        float4 AddUpdate = add_update_to_destination ? input_errors[elem_id] : make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        val.x += AddUpdate.x;
        val.y += AddUpdate.y;
        val.z += AddUpdate.z;
        val.w += AddUpdate.w;

        input_errors[elem_id] = val;
    }
}