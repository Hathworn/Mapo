#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "rectified_linear_backprop_upd_kernel.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float4 __restrict *input_errors = NULL;
hipMalloc(&input_errors, XSIZE*YSIZE);
const float4 __restrict *output_errors = NULL;
hipMalloc(&output_errors, XSIZE*YSIZE);
const uint4 __restrict *bits_buffer = NULL;
hipMalloc(&bits_buffer, XSIZE*YSIZE);
float negative_slope = 1;
bool add_update_to_destination = 1;
int elem_count = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
rectified_linear_backprop_upd_kernel<<<gridBlock, threadBlock>>>(input_errors,output_errors,bits_buffer,negative_slope,add_update_to_destination,elem_count);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
rectified_linear_backprop_upd_kernel<<<gridBlock, threadBlock>>>(input_errors,output_errors,bits_buffer,negative_slope,add_update_to_destination,elem_count);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
rectified_linear_backprop_upd_kernel<<<gridBlock, threadBlock>>>(input_errors,output_errors,bits_buffer,negative_slope,add_update_to_destination,elem_count);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}