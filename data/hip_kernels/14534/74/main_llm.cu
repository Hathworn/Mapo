#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "apply_gradient_with_weight_decay_util_kernel_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    const float2 __restrict *gradient = NULL;
hipMalloc(&gradient, XSIZE*YSIZE);
const float2 __restrict *learning_rates = NULL;
hipMalloc(&learning_rates, XSIZE*YSIZE);
float2 __restrict *weights = NULL;
hipMalloc(&weights, XSIZE*YSIZE);
float weight_decay = 1;
int elem_count = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
apply_gradient_with_weight_decay_util_kernel<<<gridBlock, threadBlock>>>(gradient,learning_rates,weights,weight_decay,elem_count);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
apply_gradient_with_weight_decay_util_kernel<<<gridBlock, threadBlock>>>(gradient,learning_rates,weights,weight_decay,elem_count);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
apply_gradient_with_weight_decay_util_kernel<<<gridBlock, threadBlock>>>(gradient,learning_rates,weights,weight_decay,elem_count);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}