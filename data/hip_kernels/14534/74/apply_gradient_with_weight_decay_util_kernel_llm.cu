#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_gradient_with_weight_decay_util_kernel(const float2 * __restrict__ gradient, const float2 * __restrict__ learning_rates, float2 * __restrict__ weights, float weight_decay, int elem_count)
{
    // Calculate unique thread index
    int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure thread is within bounds
    if (elem_id < elem_count)
    {
        // Load data from global memory to registers
        float2 lr = learning_rates[elem_id];
        float2 current_weight = weights[elem_id];
        float2 grad = gradient[elem_id];

        // Perform computation on registers
        weights[elem_id].x = current_weight.x + lr.x * (grad.x - weight_decay * current_weight.x);
        weights[elem_id].y = current_weight.y + lr.y * (grad.y - weight_decay * current_weight.y);
    }
}