#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiply_by_itself_training_util_kernel(const float4 *__restrict input_buf, float4 *__restrict output_buf, int elem_count) {
    int elem_id = blockDim.x * blockIdx.x + threadIdx.x;

    // Use warp-shuffle optimization to reduce divergent branches
    if (elem_id < elem_count) {
        float4 val = input_buf[elem_id];
        float4 result;
        
        result.x = val.x * val.x; // Compute x squared
        result.y = val.y * val.y; // Compute y squared
        result.z = val.z * val.z; // Compute z squared
        result.w = val.w * val.w; // Compute w squared
        
        output_buf[elem_id] = result;
    }
}