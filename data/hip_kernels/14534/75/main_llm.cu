#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "multiply_by_itself_training_util_kernel_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    const float4 __restrict *input_buf = NULL;
hipMalloc(&input_buf, XSIZE*YSIZE);
float4 __restrict *output_buf = NULL;
hipMalloc(&output_buf, XSIZE*YSIZE);
int elem_count = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
multiply_by_itself_training_util_kernel<<<gridBlock, threadBlock>>>(input_buf,output_buf,elem_count);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
multiply_by_itself_training_util_kernel<<<gridBlock, threadBlock>>>(input_buf,output_buf,elem_count);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
multiply_by_itself_training_util_kernel<<<gridBlock, threadBlock>>>(input_buf,output_buf,elem_count);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}