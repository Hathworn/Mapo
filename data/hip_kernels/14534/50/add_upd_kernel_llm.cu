#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_upd_kernel(float4 * __restrict__ output_buf, const float4 * __restrict__ input_buf1, const float4 * __restrict__ input_buf2, float alpha1, float alpha2, int elem_count)
{
    int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure we only process the elements within bounds
    if (elem_id < elem_count)
    {
        // Load inputs
        float4 val1 = __ldg(&input_buf1[elem_id]);
        float4 val2 = __ldg(&input_buf2[elem_id]);
        
        // Compute output value using fused multiply-add for potential optimization
        val1.x = fmaf(val1.x, alpha1, val2.x * alpha2);
        val1.y = fmaf(val1.y, alpha1, val2.y * alpha2);
        val1.z = fmaf(val1.z, alpha1, val2.z * alpha2);
        val1.w = fmaf(val1.w, alpha1, val2.w * alpha2);
        
        // Store result
        output_buf[elem_id] = val1;
    }
}