#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rectified_linear_upd_kernel( const float4 * __restrict input, float4 * __restrict output, uint4 * __restrict bits_buffer, float negative_slope, int elem_count)
{
    int elem_id = blockDim.x * blockIdx.x + threadIdx.x;

    if (elem_id < elem_count)
    {
        float4 val = input[elem_id]; // Directly loading input into val
        uint4 bits;

        // Using __ballot_sync for both CUDA 9 and above for consistency
        bits.x = __ballot_sync(0xFFFFFFFF, val.x < 0.0F ? 0 : 1);
        bits.y = __ballot_sync(0xFFFFFFFF, val.y < 0.0F ? 0 : 1);
        bits.z = __ballot_sync(0xFFFFFFFF, val.z < 0.0F ? 0 : 1);
        bits.w = __ballot_sync(0xFFFFFFFF, val.w < 0.0F ? 0 : 1);

        int lane_id = elem_id & 31;
        if (lane_id == 0)
            bits_buffer[elem_id >> 5] = bits;

        // Inline negative slope application to reduce branching
        val.x = val.x < 0.0F ? val.x * negative_slope : val.x;
        val.y = val.y < 0.0F ? val.y * negative_slope : val.y;
        val.z = val.z < 0.0F ? val.z * negative_slope : val.z;
        val.w = val.w < 0.0F ? val.w * negative_slope : val.w;

        output[elem_id] = val;
    }
}