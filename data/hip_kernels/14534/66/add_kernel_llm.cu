#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(float4 * __restrict output_buf, const float4 * __restrict input_buf1, const float4 * __restrict input_buf2, float alpha1, float alpha2, int elem_count)
{
    int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
    if (elem_id >= elem_count) return; // Early exit for threads outside range
    
    float4 val1 = input_buf1[elem_id];
    float4 val2 = input_buf2[elem_id];

    // Compute and assign in one step to reduce latency
    output_buf[elem_id] = make_float4(
        val1.x * alpha1 + val2.x * alpha2,
        val1.y * alpha1 + val2.y * alpha2,
        val1.z * alpha1 + val2.z * alpha2,
        val1.w * alpha1 + val2.w * alpha2
    );
}