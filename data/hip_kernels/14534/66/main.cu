#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "add_kernel.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float4 __restrict *output_buf = NULL;
hipMalloc(&output_buf, XSIZE*YSIZE);
const float4 __restrict *input_buf1 = NULL;
hipMalloc(&input_buf1, XSIZE*YSIZE);
const float4 __restrict *input_buf2 = NULL;
hipMalloc(&input_buf2, XSIZE*YSIZE);
float alpha1 = 2;
float alpha2 = 2;
int elem_count = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
add_kernel<<<gridBlock, threadBlock>>>(output_buf,input_buf1,input_buf2,alpha1,alpha2,elem_count);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
add_kernel<<<gridBlock, threadBlock>>>(output_buf,input_buf1,input_buf2,alpha1,alpha2,elem_count);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
add_kernel<<<gridBlock, threadBlock>>>(output_buf,input_buf1,input_buf2,alpha1,alpha2,elem_count);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}