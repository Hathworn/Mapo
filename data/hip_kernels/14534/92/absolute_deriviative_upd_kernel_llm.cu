#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access and reduce branching
__global__ void absolute_deriviative_upd_kernel( float4 * __restrict input_errors, const float4 * __restrict output_errors, const float4 * __restrict input_neurons, bool add_update_to_destination, int elem_count)
{
    int elem_id = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    if (elem_id < elem_count)
    {
        // Load data only once to optimize memory access
        float4 inp = input_neurons[elem_id];
        float4 current_error = output_errors[elem_id];
        
        // Use ternary operator to eliminate branching
        current_error.x = inp.x < 0.0f ? -current_error.x : current_error.x;
        current_error.y = inp.y < 0.0f ? -current_error.y : current_error.y;
        current_error.z = inp.z < 0.0f ? -current_error.z : current_error.z;
        current_error.w = inp.w < 0.0f ? -current_error.w : current_error.w;

        if (add_update_to_destination)
        {
            float4 current_dst = input_errors[elem_id];
            current_error.x += current_dst.x;
            current_error.y += current_dst.y;
            current_error.z += current_dst.z;
            current_error.w += current_dst.w;
        }
        
        // Write back the updated error
        input_errors[elem_id] = current_error;
    }
}