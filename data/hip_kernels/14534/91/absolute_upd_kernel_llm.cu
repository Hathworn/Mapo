#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void absolute_upd_kernel(const float4* __restrict__ input, float4* __restrict__ output, int elem_count)
{
    // Calculate element index using faster multiplication with constants
    int elem_id = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    // Check bounds and perform computation if within element count
    if (elem_id < elem_count)
    {
        float4 val = input[elem_id];

        // Optimize absolute value computation
        val.x = fabsf(val.x);
        val.y = fabsf(val.y);
        val.z = fabsf(val.z);
        val.w = fabsf(val.w);

        output[elem_id] = val;
    }
}