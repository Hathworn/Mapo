#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_weight_decay_util_kernel(
    const float4* __restrict__ learning_rates, 
    float4* __restrict__ weights, 
    float weight_decay, 
    int elem_count) 
{
    int elem_id = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (elem_id < elem_count) 
    {
        // Load data from global memory only once
        float4 val = __ldg(&learning_rates[elem_id]);
        float4 current_weight = __ldg(&weights[elem_id]);

        // Calculate decay factor and update weights
        float decay_factor_x = 1.0F - val.x * weight_decay;
        float decay_factor_y = 1.0F - val.y * weight_decay;
        float decay_factor_z = 1.0F - val.z * weight_decay;
        float decay_factor_w = 1.0F - val.w * weight_decay;

        current_weight.x *= decay_factor_x;
        current_weight.y *= decay_factor_y;
        current_weight.z *= decay_factor_z;
        current_weight.w *= decay_factor_w;

        // Store the result back to global memory
        weights[elem_id] = current_weight;
    }
}