#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        // Use local variable to optimize memory access
        float in_val = indata[tid];
        float out_grad = outgrad[tid];
        ingrad[tid] = in_val > 0 ? out_grad : 0;
    }
}