#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Utilize all threads in the block to set value instead of checking threadIdx.x
__global__ void setValue(float *data, int idx, float value) {
    if (blockIdx.x * blockDim.x + threadIdx.x == idx) {
        data[idx] = value;
    }
}