#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getValue(float *outdata, float *indata) {
    // Use thread index to ensure data parallelism
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Conditional assignment with efficient memory access
    if (idx == 0) {
        outdata[idx] = (indata[0] == 0) ? 3.0f : 2.0f;
    }
}