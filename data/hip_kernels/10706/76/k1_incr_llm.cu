#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void k1_incr(float *data, float val) {
    // Calculate the global thread id
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use global thread id to enable processing of larger arrays
    data[idx] += val;
}