#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void longKernel(float *data, int N, float value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int stride = gridDim.x * blockDim.x; // Calculate total number of threads in the grid
    for (int i = idx; i < N; i += stride) { // Spread work across multiple threads
        data[i] += value;
    }
}