#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getValue(float *outdata, float *indata) {
    // Use shared memory to reduce global memory access latency
    __shared__ float sharedData;
    if (threadIdx.x == 0) {
        sharedData = indata[0]; // Load data into shared memory by the first thread
    }
    __syncthreads(); // Ensure shared memory is loaded before use
    
    if (threadIdx.x == 0) {
        outdata[0] = sharedData + 3.0f; // Use shared memory data
    }
}