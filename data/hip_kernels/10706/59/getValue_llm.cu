#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getValue(float4 *outdata, float *indata) {
    // Reordering memory access to enhance coalescing
    float4 my4 = make_float4(indata[0], indata[1], indata[2], indata[3]);
    outdata[0] = my4;
}