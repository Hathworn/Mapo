#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getValue(float *outdata, float *indata) {
    // Simplify conditional logic by using ternary operator directly
    *outdata = *indata == 0 ? 3.0f : 2.0f;
}