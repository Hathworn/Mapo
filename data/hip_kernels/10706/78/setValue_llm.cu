#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setValue(float *data, int idx, float value) {
    // Use warp size to parallelize the operation
    int laneID = threadIdx.x % warpSize;
    if (laneID == 0) {
        atomicExch(&data[idx], value);
    }
}