#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void longKernel(float *data, int N, float value) {
    // Compute unique index for each thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (idx < N) {
        // Update data at the computed index
        data[idx] += value;
    }
}