#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Simplified kernel function using ternary operator correctly
__global__ void getValue(float *outdata, float *indata) {
    outdata[0] = (*indata == 0.0f) ? 3.0f : 2.0f;
}