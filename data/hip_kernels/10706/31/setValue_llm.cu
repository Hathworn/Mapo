#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setValue(float *data, int idx, float value) {
    // Use blockIdx.x and blockDim.x for control instead of checking threadIdx.x for efficiency
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid == 0) {
        data[idx] = value;
    }
}