#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized to support parallelism and more efficient memory access
__global__ void getValue(float *outdata, float *indata) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Calculate global index
    if (idx < 1) {  // Boundary check for the array size
        outdata[idx] = indata[idx] + 3.0f;
    }
}