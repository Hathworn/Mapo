#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel
__global__ void main_set(float *data, float val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Compute global index
    data[idx] = val; // Assign value using global index
}