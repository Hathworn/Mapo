#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inttofloat(float *out, int *in) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global index
    out[idx] = __int_as_float(in[idx]); // Perform int to float conversion using HIP intrinsic function
}