#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void floattoint(int *out, float *in) {
    // Optimize by ensuring each thread processes one element
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    out[idx] = __float_as_int(in[idx]); // Use intrinsic function for conversion
}