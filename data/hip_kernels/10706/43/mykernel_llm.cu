#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mykernel(float *d1, float *d2, float *d3, float *d4, float *d5) {
    // Use thread 0 in block 0 to perform initialization once
    if(blockIdx.x == 0 && threadIdx.x == 0) {
        float val = 123.0f; // Assign the value once to reduce redundant operations
        d1[0] = val;
        d2[0] = val;
        d3[0] = val;
        d4[0] = val;
        d5[0] = val;
    }
}