#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void k2_mul(float *data, float val) {
    // Calculate global index to allow for flexible grid configuration
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure threads don't access out-of-bound memory
    if (idx < N) {
        data[idx] *= val;
    }
}
```
Please ensure to define `N` before using this code, it should represent the total number of elements in the `data` array.