#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setValue(float *data, int idx, float value) {
    // Use all threads for potential reduction in latency
    if(blockIdx.x * blockDim.x + threadIdx.x == 0) {
        data[idx] = value;
    }
}