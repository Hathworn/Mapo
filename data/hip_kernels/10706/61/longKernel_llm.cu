#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void longKernel(float *data, int N, float value) {
    // Get global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bounds to prevent overflow
    if (idx < N) {
        data[idx] += value;
    }
}
```
