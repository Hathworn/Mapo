#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void longKernel(float *data, int N, float value) {
    // Optimize kernel by using thread indexing
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        data[idx] += value;
    }
}