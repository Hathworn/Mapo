#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void incrValue(float *data, int idx, float value) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        atomicAdd(&data[idx], value); // Use atomic operation for safe value increment
    }
}