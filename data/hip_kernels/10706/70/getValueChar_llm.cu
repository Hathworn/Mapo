#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getValueChar(char *outdata, char *indata) {
    // Calculate global index for the thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure we don't access out of bounds
    if (idx == 0) {
        outdata[idx] = indata[idx] + 3;
    }
}