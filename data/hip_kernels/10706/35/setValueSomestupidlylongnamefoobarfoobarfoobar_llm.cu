#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void setValueSomestupidlylongnamefoobarfoobarfoobarhaha(float *data, int idx, float value) {
    data[idx] = value;
}

__device__ float bar(float a, float b) {
    return a + b;
}

__global__ void setValueSomestupidlylongnamefoobarfoobarfoobar(float *data, int idx, float value) {
    // Optimize by handling multiple elements within a block
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid == 0) {
        setValueSomestupidlylongnamefoobarfoobarfoobarhaha(data, idx, value);
    }
}