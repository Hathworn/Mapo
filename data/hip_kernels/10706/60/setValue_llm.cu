#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setValue(float *data, int idx, float value) {
    // Use a block-level check for setting value to prevent warp divergence
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        data[idx] = value;
    }
}