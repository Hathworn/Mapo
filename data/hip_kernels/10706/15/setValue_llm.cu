#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

__global__ void setValue(float *data, int idx, float value) {
    // Assign only if the target index is within bounds and is accessed by the first thread
    if(threadIdx.x == 0 && idx < gridDim.x * blockDim.x) {
        data[idx] = value;
    }
}