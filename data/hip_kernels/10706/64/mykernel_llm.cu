#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mykernel(int *int1, float *f1, int *int2) {
    // Utilize shared memory for efficient data access
    __shared__ float sharedFloat;
    __shared__ int sharedInt;

    // Convert int to float and store in shared memory
    if (threadIdx.x == 0) {
        sharedFloat = __int_as_float(int1[0]);
    }
    __syncthreads();

    // Fetch from shared memory
    f1[0] = sharedFloat;

    // Convert float to int and store in shared memory
    if (threadIdx.x == 0) {
        sharedInt = __float_as_int(sharedFloat);
    }
    __syncthreads();

    // Fetch from shared memory
    int2[0] = sharedInt;
}