#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void wipe(int *buffer, int length) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // Global thread index
    int totalThreads = blockDim.x * gridDim.x;       // Total number of threads
    length >>= 5;
    for(int i = tid; i < length; i += totalThreads) { // Process data in parallel across threads
        buffer[(i << 5) + threadIdx.x] = -1;
    }
}