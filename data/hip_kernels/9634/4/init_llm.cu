#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(float* xbar, float* xcur, float* xn, float* y1, float* y2, float* img, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < w && y < h) {
        int baseIdx = x + w * y;
        for (int z = 0; z < nc; z++) {
            int i = baseIdx + w * h * z;
            float val = img[i];
            xbar[i] = val; // Initialize xbar
            xn[i] = val;   // Initialize xn
            xcur[i] = val; // Initialize xcur
            y1[i] = 0.f;   // Initialize y1
            y2[i] = 0.f;   // Initialize y2
        }
    }
}