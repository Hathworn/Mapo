#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void primal(float *y1, float *y2, float *xbar, float sigma, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < w && y < h) {
        // Unrolling the loop for improved performance
        int i;
        float x1, x2, val, norm;

        i = x + w * y;

        #pragma unroll
        for (int z = 0; z < nc; z++) {
            val = xbar[i];
            x1 = (x + 1 < w) ? (xbar[i + 1] - val) : 0.f;
            x2 = (y + 1 < h) ? (xbar[i + w] - val) : 0.f;

            x1 = y1[i] + sigma * x1;
            x2 = y2[i] + sigma * x2;

            norm = rsqrtf(fmaxf(1.f, x1 * x1 + x2 * x2)); // Use rsqrtf for reciprocal of square root

            y1[i] = x1 * norm;
            y2[i] = x2 * norm;

            i += w * h; // Increment i for next color channel
        }
    }
}