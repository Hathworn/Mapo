#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dual(float* xn, float* xcur, float* y1, float* y2, float* img, float tau, float lambda, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < w && y < h) {
        float d1, d2, val, value;
        float factor = tau * lambda;

        for (int z = 0; z < nc; z++) {
            int i = x + w * y + w * h * z;

            // Compute d1 and d2 with boundary checks
            d1 = (x + 1 < w ? y1[i] : 0.f) - (x > 0 ? y1[(x - 1) + w * y + w * h * z] : 0.f);
            d2 = (y + 1 < h ? y2[i] : 0.f) - (y > 0 ? y2[x + w * (y - 1) + w * h * z] : 0.f);

            // Temporal variable to store the calculated val
            val = xcur[i] + tau * (d1 + d2);
            value = val - img[i];

            // Simplify the value decision checks
            if (fabs(value) > factor) {
                xn[i] = val - copysignf(factor, value);  // Use copysignf for concise factor correction
            } else {
                xn[i] = img[i];
            }
        }
    }
}