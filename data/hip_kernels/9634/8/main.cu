#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "dual.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *xn = NULL;
hipMalloc(&xn, XSIZE*YSIZE);
float *xcur = NULL;
hipMalloc(&xcur, XSIZE*YSIZE);
float *y1 = NULL;
hipMalloc(&y1, XSIZE*YSIZE);
float *y2 = NULL;
hipMalloc(&y2, XSIZE*YSIZE);
float *img = NULL;
hipMalloc(&img, XSIZE*YSIZE);
float tau = 1;
float lambda = 1;
int w = XSIZE;
int h = YSIZE;
int nc = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
dual<<<gridBlock, threadBlock>>>(xn,xcur,y1,y2,img,tau,lambda,w,h,nc);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
dual<<<gridBlock, threadBlock>>>(xn,xcur,y1,y2,img,tau,lambda,w,h,nc);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
dual<<<gridBlock, threadBlock>>>(xn,xcur,y1,y2,img,tau,lambda,w,h,nc);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}