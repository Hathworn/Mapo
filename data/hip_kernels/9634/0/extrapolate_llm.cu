#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void extrapolate(float* xbar, float* xcur, float* xn, float theta, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < w && y < h) {
        int baseIndex = x + w * y; // Precompute 2D index

        for (int z = 0; z < nc; z++) {
            int i = baseIndex + w * h * z; // Compute 3D index
            float currentXn = xn[i]; // Cache xn[i] value
            xbar[i] = currentXn + theta * (currentXn - xcur[i]); // Optimize calculation
            xcur[i] = currentXn;
        }
    }
}