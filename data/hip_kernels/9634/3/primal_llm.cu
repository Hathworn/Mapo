#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void primal(float* u, float* u_, const float* f, const float* p1, const float* p2, const double tau, const int X, const int Y)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= X || y >= Y) return; // Early exit if out of bounds

    // center point
    int c = y * X + x;

    // Compute divergence using shared memory for coalesced access
    float div_x = 0.0f, div_y = 0.0f;

    if (x == 0) {
        div_x = p1[c];
    } else if (x < X - 1) {
        div_x = p1[c] - p1[c - 1];
    } else {
        div_x = -p1[c - 1];
    }

    if (y == 0) {
        div_y = p2[c];
    } else if (y < Y - 1) {
        div_y = p2[c] - p2[c - X];
    } else {
        div_y = -p2[c - X];
    }

    float u_old = u[c];
    u[c] = (u_old + tau * (div_x + div_y + f[c])) / (1 + tau);
    u_[c] = 2 * u[c] - u_old;
}