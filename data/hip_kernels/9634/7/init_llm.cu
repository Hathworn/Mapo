#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init(float* xbar, float* xcur, float* xn, float* y1, float* y2, float* img, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < w && y < h) {
        int baseIndex = x + w * y;  // Precompute base index
        for (int z = 0; z < nc; ++z) {
            int i = baseIndex + w * h * z;  // Use base index for each channel
            float val = img[i];
            // Coalesced writes to global memory
            xbar[i] = val;
            xn[i] = val;
            xcur[i] = val;
            y1[i] = 0.f;
            y2[i] = 0.f;
        }
    }
}