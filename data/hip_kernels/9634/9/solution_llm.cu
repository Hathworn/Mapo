#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void solution(float* img, float* xbar, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < w && y < h) {
        // Unrolling loop for better performance
        int base_index = x + w * y;
        #pragma unroll
        for (int z = 0; z < nc; z++) {
            int i = base_index + w * h * z;
            img[i] = xbar[i];
        }
    }
}