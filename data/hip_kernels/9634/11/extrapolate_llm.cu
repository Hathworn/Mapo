#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void extrapolate(float* xbar, float* xcur, float* xn, float theta, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    int z, i;

    // Check boundary conditions for x and y
    if (x < w && y < h) {
        // Loop unrolling optimization for better performance
        for (z = 0; z < nc; z++) {
            i = x + w * (y + h * z);
            float xn_val = xn[i];  // Cache xn value to reduce memory access
            float diff = xn_val - xcur[i];  // Compute the difference once
            xbar[i] = xn_val + theta * diff;  // Use cached values
            xcur[i] = xn_val;
        }
    }
}