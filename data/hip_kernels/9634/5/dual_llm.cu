#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dual(float* p1, float* p2, const float* u_, const double lambda, const double sigma, const int X, const int Y)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within bounds
    if (x >= X || y >= Y) return;

    // Center point
    int c = y * X + x;

    float nabla_x = 0.0f;
    float nabla_y = 0.0f;

    // Compute differences if within valid range
    if (x < X - 1) nabla_x = u_[c + 1] - u_[c];
    if (y < Y - 1) nabla_y = u_[c + X] - u_[c];

    // Update p1 and p2
    p1[c] += sigma * nabla_x;
    p2[c] += sigma * nabla_y;

    // Normalization
    float denom = fmaxf(1.0f, sqrtf(p1[c] * p1[c] + p2[c] * p2[c]) / lambda);
    p1[c] /= denom;
    p2[c] /= denom;
}