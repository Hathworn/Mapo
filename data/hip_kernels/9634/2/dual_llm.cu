#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dual(float* xn, float* xcur, float* y1, float* y2, float* img, float tau, float lambda, int w, int h, int nc) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;

    if (x < w && y < h) {
        int i = x + w * y;  // Calculate the base index once for all channels

        // Separate computations into variables outside the loop
        float invFactor = 1.f / (1.f + tau * lambda);
        float correctionTerm;
        float y1Left, y1Right, y2Top, y2Bottom;

        for (int z = 0; z < nc; z++) {
            int baseIndex = i + w * h * z;  // Calculate the index for each channel

            // Use ternary operators to simplify boundary condition checks
            y1Right = (x+1 < w) ? y1[baseIndex] : 0.f;
            y1Left = (x > 0) ? y1[baseIndex - 1] : 0.f;
            y2Bottom = (y+1 < h) ? y2[baseIndex] : 0.f;
            y2Top = (y > 0) ? y2[baseIndex - w] : 0.f;

            // Perform calculations
            correctionTerm = tau * ((y1Right - y1Left) + (y2Bottom - y2Top));
            xn[baseIndex] = (xcur[baseIndex] + correctionTerm + tau * lambda * img[baseIndex]) * invFactor;
        }
    }
}