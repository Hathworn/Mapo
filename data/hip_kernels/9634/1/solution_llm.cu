#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void solution(float* img, float* xbar, int w, int h, int nc) {
    // Calculate global thread index
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    int y = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (x < w && y < h) {
        // Calculate base index for thread
        int baseIdx = x + w * y;
        for (int z = 0; z < nc; z++) {
            // Directly compute global index using base index
            int i = baseIdx + w * h * z;
            img[i] = xbar[i];
        }
    }
}