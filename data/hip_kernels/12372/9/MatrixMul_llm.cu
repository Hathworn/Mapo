#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMul(float *Md, float *Nd, float *Pd, const int WIDTH) {
    // calculate thread id
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

    // Use shared memory to optimize memory access
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    float Pvalue = 0;

    // Loop through tiles
    for (int m = 0; m < WIDTH / TILE_WIDTH; ++m) {
        // Load data into shared memory with coalesced access
        Mds[threadIdx.y][threadIdx.x] = Md[row * WIDTH + (m * TILE_WIDTH + threadIdx.x)];
        Nds[threadIdx.y][threadIdx.x] = Nd[(m * TILE_WIDTH + threadIdx.y) * WIDTH + col];
        
        __syncthreads(); // Synchronize to ensure data is loaded

        // Perform the computation for a tile
        for (int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        }

        __syncthreads(); // Synchronize to prevent race conditions
    }

    // Write the result to the matrix P
    Pd[row * WIDTH + col] = Pvalue;
}