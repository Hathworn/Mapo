#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multMatriz(float *da, float *db, float *dc, int num) {
    // Utilize shared memory to reduce global memory accesses.
    __shared__ float tileA[32][32];
    __shared__ float tileB[32][32];

    float sum = 0.0f;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    // Loop over tiles.
    for (int m = 0; m < (num + 31) / 32; ++m) {
        // Load a tile from matrix A and B into shared memory.
        if (row < num && m * 32 + threadIdx.x < num)
            tileA[threadIdx.y][threadIdx.x] = da[row * num + m * 32 + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < num && m * 32 + threadIdx.y < num)
            tileB[threadIdx.y][threadIdx.x] = db[(m * 32 + threadIdx.y) * num + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        // Compute partial product for the current tile.
        for (int k = 0; k < 32; ++k) {
            sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }
        __syncthreads();
    }

    // Write the result to the output matrix.
    if (row < num && col < num) {
        dc[row * num + col] = sum;
    }
}