#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multMatriz(float *da, float *db, float *dc, int num) {
    // Calculate unique global thread row and column indices
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < num && j < num) { // Ensure indices are within bounds
        float sum = 0;
        for (unsigned int k = 0; k < num; k++) {
            sum += da[i * num + k] * db[k * num + j];
        }
        dc[i * num + j] = sum;
    }
}