#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void suma(int a, int b, int *c) {
    // Using blockIdx.x allows for modular use in grid-based parallelism
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        *c = a + b; // Perform the operation within a conditional block, ensuring only one thread executes it.
    }
}