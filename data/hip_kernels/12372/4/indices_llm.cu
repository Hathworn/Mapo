#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void indices() {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // Precompute block and thread sizes to avoid multiple accesses
    int blockDimY = blockDim.y;
    int threadX = threadIdx.x;
    int blockIdxX = blockIdx.x;
    int blockDimX = blockDim.x;
    
    // Store sqrt result in a variable to avoid duplicate computation
    double idSqrt = sqrt((double)id);

    // Optimize printf by reusing computed variables
    printf("blockdimy: %d  threadx: %d  Blockidx: %d  blockdimx: %d id:  %d raiz: %f\n",
           blockDimY, threadX, blockIdxX, blockDimX, id, idSqrt);

    __syncthreads();
}