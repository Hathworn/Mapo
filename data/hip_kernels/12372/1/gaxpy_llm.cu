#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpy(double *y, double *a, double *x, int m, int n) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    if (bid < m) {
        // Precompute index to reduce repetitive computation
        int index = bid * n + tid;
        if (tid < n) {
            // Compute the dot product directly and store locally
            dots_s[index] = a[index] * x[tid];
        }
        __syncthreads(); // Synchronize threads to ensure memory consistency

        // Perform reduction using a single thread block to improve performance
        if (tid == 0) {
            double sum = 0.0;
            for (int i = 0; i < n; i++) {
                sum += dots_s[bid * n + i];
            }
            y[bid] = sum; // Store the result in y
        }
    }
}