#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpymm(double *y, double *a, double *b, int m, int n, int p) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    if (bid < m) {
        // Use a single loop to handle multiplication and accumulation
        for (int c = 0; c < p; c++) {
            double sum = 0.0;  // Initialize accumulator
            for (int i = 0; i < n; i++) {
                sum += a[bid * n + i] * b[i * p + c];
            }
            // Store the computed result
            if (tid == 0) {
                y[bid * p + c] = sum;
            }
        }
    }
}