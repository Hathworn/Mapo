#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_example(int *c, const int *a, const int *b)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform operation if within bounds
    if (i < gridDim.x * blockDim.x) {
        c[i] = a[i] + b[i];
    }
}