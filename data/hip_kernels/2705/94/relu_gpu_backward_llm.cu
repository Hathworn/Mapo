#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    // Use shared memory for faster access
    extern __shared__ float s_indata[];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int lane = threadIdx.x;

    // Load data into shared memory
    if (tid < N) {
        s_indata[lane] = indata[tid];
        __syncthreads(); // Synchronize to ensure all data is loaded

        // Efficient calculation with shared memory
        ingrad[tid] = s_indata[lane] > 0 ? outgrad[tid] : 0;
    }
}