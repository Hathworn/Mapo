#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for loop execution
    for (int i = tid; i < N; i += stride) { // Loop over data with stride
        out[i] = in[i] > 0 ? in[i] : 0; // Set output using relu condition
    }
}