#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *a, size_t N)
{
    int tid = threadIdx.x;
    __shared__ float s[BS];
    float sum = 0.0f;
    
    for (int ib = blockIdx.x; ib < (N + BS - 1) / BS; ib += gridDim.x) { // Use grid-stride loop
        int off = ib * BS + tid;
        
        if (off < N) { // Check bounds before accessing
            s[tid] = a[off];
        } else {
            s[tid] = 0.0f; // Avoid use of invalid memory
        }
        
        __syncthreads(); // Ensure all loads to shared memory complete
        
        // Reduce in shared memory
        for (int skip = BS / 2; skip > 0; skip >>= 1) {
            if (tid < skip) {
                s[tid] += s[tid + skip];
            }
            __syncthreads(); // Sync after each reduction step
        }
        
        sum += s[0]; // Only thread 0 has the reduced sum
    }
    
    if (tid == 0) { // Only thread 0 writes to the output
        atomicAdd(&a[0], sum);
    }
}