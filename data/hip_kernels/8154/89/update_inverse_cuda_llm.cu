#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void update_inverse_cuda (float *Ainv, float *u, int N, int rowstride, int k)
{
    __shared__ float A_k[NMAX], u_shared[NMAX], Ainv_u[NMAX], Ainv_shared[NMAX];
    
    // Load A_k and u_shared into shared memory
    int tid = threadIdx.x;
    A_k[tid] = Ainv[k * rowstride + tid];
    u_shared[tid] = u[tid];
    
    // Compute the k'th element of Ainv_u
    Ainv_u[tid] = u_shared[tid] * A_k[tid];
    __syncthreads();
    
    // Efficient reduction to compute sum into Ainv_u[0]
    for (int stride = N >> 1; stride > 0; stride >>= 1) {
        if (tid < stride) {
            Ainv_u[tid] += Ainv_u[tid + stride]; // Sum partial results
        }
        __syncthreads();
    }
    
    float prefact = -1.0f / (1.0f + Ainv_u[0]);
    
    // Update Ainv for each row
    for (int row = 0; row < N; row++) {
        Ainv_shared[tid] = Ainv[row * rowstride + tid];
        __syncthreads();
        
        Ainv_u[tid] = u_shared[tid] * Ainv_shared[tid];
        
        // Efficient reduction to compute sum into Ainv_u[0]
        for (int stride = N >> 1; stride > 0; stride >>= 1) {
            if (tid < stride) {
                Ainv_u[tid] += Ainv_u[tid + stride]; // Sum partial results
            }
            __syncthreads();
        }
        
        // Update Ainv[row]
        Ainv[row * rowstride + tid] = Ainv_shared[tid] + prefact * Ainv_u[0] * A_k[tid];
        __syncthreads();
    }
}