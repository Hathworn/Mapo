#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FloatDivByConstant(float *A, float constant)
{
    // Calculate unique global thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x; 

    // Grid stride loop for larger arrays
    unsigned int stride = blockDim.x * gridDim.x;
    for (unsigned int i = tid; i < N; i += stride) 
    {
        A[i] = A[i] / constant; // Division operation
    }
}