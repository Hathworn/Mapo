#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FloatMul(float *A, float *B, float *C)
{
    // Calculate the global thread index directly
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Efficiently multiply arrays A and B
    C[i] = A[i] * B[i];
}