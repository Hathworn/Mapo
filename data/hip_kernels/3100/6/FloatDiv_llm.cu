#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FloatDiv(float *A, float *B, float *C)
{
    unsigned int i = blockIdx.x * blockDim.x * blockDim.y * blockDim.z  // Simplified index calculation
                    + blockIdx.y * blockDim.y * blockDim.z
                    + blockIdx.z * blockDim.z
                    + threadIdx.z * blockDim.y * blockDim.x 
                    + threadIdx.y * blockDim.x 
                    + threadIdx.x;

    // Perform boundary check
    unsigned int totalThreads = gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;
    if (i < totalThreads) {
        C[i] = (B[i] != 0) ? A[i] / B[i] : 0;
    }
}