#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(float* d_out, float* d_in) {
    // Compute the global index of the thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Compute squared value only if global index is within bounds
    if (idx < N) {
        float f = d_in[idx];
        d_out[idx] = f * f;
    }
}