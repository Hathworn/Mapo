#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Utilizing shared memory to reduce redundancy
__global__ void hello()
{
    // Cache block index in shared memory for faster access
    __shared__ int cachedBlockIdx;
    if (threadIdx.x == 0) {
        cachedBlockIdx = blockIdx.x; // Assign block index to shared memory
    }
    __syncthreads(); // Ensure all threads see the updated block index

    printf("Hello world! I'm a thread in block %d\n", cachedBlockIdx);
}