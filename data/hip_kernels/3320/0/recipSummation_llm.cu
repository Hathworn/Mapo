#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void recipSummation(double* data, double* recip, int len)
{
    // Calculate global thread index
    const int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    if (i < len) {
        const int j = 2 * i;

        // Multiply by reciprocal for both elements consecutively
        double recip_val = recip[i];
        data[j]     *= recip_val;
        data[j + 1] *= recip_val;
    }
}