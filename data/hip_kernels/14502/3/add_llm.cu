#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c, int *d, int *e, int *f) {
    int valA = *a; // Load values from global memory once for efficiency
    int valB = *b;
    
    *c = valA + valB; // Compute sum
    *d = valA - valB; // Compute difference
    *e = valA * valB; // Compute product
    
    if (valB != 0) { // Check to avoid division by zero
        *f = valA / valB; // Compute quotient
    }
}