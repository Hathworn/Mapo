#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void trapz_kernel(float* y, float* x, float* auc, int num_selected) {
    // Use shared memory to reduce atomic operations on global memory
    __shared__ float s_auc_local[256]; // Adjust size based on blockDim.x
    int tid = threadIdx.x;
    s_auc_local[tid] = 0.0f;
    __syncthreads();

    int gid_base = blockIdx.x * blockDim.x + threadIdx.x;
    for (int gid = gid_base; gid < num_selected - 1; gid += blockDim.x * gridDim.x) {
        float a = x[gid];
        float b = x[gid + 1];
        float fa = y[gid];
        float fb = y[gid + 1];
        float area = (b - a) * (fa + fb) / 2.0f;
        if (gid == 0) {
            area += (a * fa / 2.0f);
        }
        // Accumulate area in shared memory
        s_auc_local[tid] += area;
    }
    __syncthreads();

    // Reduce within block and write result to global memory
    if (tid == 0) {
        float block_sum = 0.0f;
        for (int i = 0; i < blockDim.x; i++) {
            block_sum += s_auc_local[i];
        }
        atomicAdd(auc, block_sum);
    }
}