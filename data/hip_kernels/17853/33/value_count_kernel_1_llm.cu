#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void value_count_kernel_1(int nnz, const size_t *hash_value_index_sort, uint32_t *new_hash_value_flag) {
    // Optimize using shared memory for reduced global memory access
    __shared__ size_t shared_hash_value[BLOCKDIM];
    
    // Calculate global thread index
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load current hash value into shared memory
    if (gid < nnz) {
        shared_hash_value[threadIdx.x] = hash_value_index_sort[gid];
    }
    
    // Synchronize to ensure all threads have loaded their data
    __syncthreads();

    // Process data once all are loaded
    if (gid < nnz) {
        size_t cur_value = shared_hash_value[threadIdx.x];
        
        // Handle first element separately
        if (gid > 0) {
            size_t former_value = (threadIdx.x == 0) ? hash_value_index_sort[gid - 1] : shared_hash_value[threadIdx.x - 1];
            new_hash_value_flag[gid] = (cur_value != former_value) ? 1 : 0;
        } else {  // gid == 0
            new_hash_value_flag[gid] = 1;
        }
    }
}