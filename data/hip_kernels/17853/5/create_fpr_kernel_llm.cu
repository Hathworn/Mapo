#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void create_fpr_kernel(float* tpr, const int* unique_index, float* fpr, int num_selected, int num_total) {
    // Precompute constants outside of loop
    const float pos_cnt = tpr[num_selected - 1];
    const float neg_cnt = num_total - pos_cnt;
    const int gid_stride = blockDim.x * gridDim.x;

    // Calculate global thread index
    const int gid_base = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride-based loop for better parallelism
    for (int gid = gid_base; gid < num_selected; gid += gid_stride) {
        const float tp = tpr[gid];
        fpr[gid] = (1.0f + unique_index[gid] - tp) / neg_cnt;
        tpr[gid] = tp / pos_cnt;
    }
}