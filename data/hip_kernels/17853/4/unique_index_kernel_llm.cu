#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unique_index_kernel(const char* flag, const int* flag_inc_sum, int* unique_index, int num_elems) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global index
    if (gid < num_elems && flag[gid] == 1) { // Check boundaries and condition
        int id = flag_inc_sum[gid] - 1; // Obtain the unique index
        unique_index[id] = gid; // Assign the global index to the unique index
    }
}