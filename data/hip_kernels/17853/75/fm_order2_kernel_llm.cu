#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fm_order2_kernel(const float* in, float* out, int batch_size, int slot_num, int emb_vec_size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;  // Use entire grid for indexing
    int total_threads = batch_size * emb_vec_size;

    if (tid < total_threads) {
        int bid = tid / emb_vec_size;
        int local_tid = tid % emb_vec_size;

        float emb_sum = 0.0f;
        float emb_sum_square = 0.0f;
        float emb_square_sum = 0.0f;
        int offset = bid * slot_num * emb_vec_size + local_tid;

        for (int i = 0; i < slot_num; i++) {
            int index = offset + i * emb_vec_size;
            float temp = in[index];
            emb_sum += temp;
            emb_square_sum += temp * temp;
        }
        emb_sum_square = emb_sum * emb_sum;

        out[tid] = 0.5f * (emb_sum_square - emb_square_sum);
    }
}