#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fm_order2_dgrad_kernel(const float* in, const float* top_grad, float* dgrad, int batch_size, int slot_num, int emb_vec_size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // Global index for parallelization

    if (tid < batch_size * emb_vec_size) { // Single condition for valid range
        int bid = tid / emb_vec_size; // Calculate batch index
        int vec_offset = tid % emb_vec_size; // Calculate vector offset
        float emb_sum = 0.0f;
        int offset = bid * slot_num * emb_vec_size + vec_offset;

        for (int i = 0; i < slot_num; i++) {
            int index = offset + i * emb_vec_size;
            emb_sum += in[index];
        }

        float tgrad = top_grad[bid * emb_vec_size + vec_offset];
        for (int i = 0; i < slot_num; i++) {
            int index = offset + i * emb_vec_size;
            dgrad[index] = tgrad * (emb_sum - in[index]);
        }
    }
}