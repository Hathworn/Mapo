#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_kernel(uint32_t hash_value_index_count_num, int embedding_vec_size, const size_t *deltaw_hash_value_index, const float *deltaw, float *hash_table_value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index

    if (idx < hash_value_index_count_num * embedding_vec_size) { // Check bounds for all thread indices
        size_t bid = idx / embedding_vec_size; // Calculate bid from global index
        size_t tid = idx % embedding_vec_size; // Calculate tid from global index

        size_t value_index = deltaw_hash_value_index[bid];
        size_t feature_index = value_index * embedding_vec_size + tid;

        hash_table_value[feature_index] += deltaw[idx];
    }
}