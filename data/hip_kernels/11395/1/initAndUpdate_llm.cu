#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initAndUpdate( float *D_oldVal, float *D_currVal, int tpoints, int nsteps )
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    if ( j < tpoints )
    {
        j += 1;
        float x = ( float )( j - 1 ) / ( tpoints - 1 );
        float sine_val = sinf(6.2831853f * x);  // Use sinf for potentially faster single precision
        D_oldVal[j] = D_currVal[j] = sine_val;
        
        if ( ( j == 1 ) || ( j  == tpoints ) )
        {
            D_currVal[j] = 0.0f;  // Use f suffix for consistency with float
        }
        else
        {
            float old = D_oldVal[j];
            float curr = D_currVal[j];
            for ( int i = 0; i < nsteps; i++ )  // Change loop to start from zero for cleaner code
            {
                float newVal = (2.0f * curr) - old + (-0.18f * curr);  // Combine constants
                old = curr;
                curr = newVal;
            }
            D_oldVal[j] = old;
            D_currVal[j] = curr;
        }
    }
}