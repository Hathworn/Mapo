#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CycleRoutineGPU(char *CurrentState, char *NextState, int X, int Dimension) {
    int sum = 0;
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < Dimension) {
        int idPlusX = id + X;
        int idMinusX = id - X;
        int row = id / X;

        // Simplified boundary conditions
        if (idPlusX < Dimension) sum += CurrentState[idPlusX]; // Down
        if (idMinusX >= 0) sum += CurrentState[idMinusX];      // Up
        if (id % X != X - 1) sum += CurrentState[id + 1];      // Right
        if (id % X != 0) sum += CurrentState[id - 1];          // Left

        // Diagonal checks within the same row boundary
        if (idPlusX < Dimension && (idPlusX % X) != X - 1) sum += CurrentState[idPlusX + 1]; // Down-Right
        if (idPlusX < Dimension && (idPlusX % X) != 0) sum += CurrentState[idPlusX - 1];     // Down-Left
        if (idMinusX >= 0 && (idMinusX % X) != X - 1) sum += CurrentState[idMinusX + 1];     // Up-Right
        if (idMinusX >= 0 && (idMinusX % X) != 0) sum += CurrentState[idMinusX - 1];         // Up-Left

        // State transition
        if (sum < 2 || sum > 3)
            NextState[id] = 0;
        else if (sum == 3)
            NextState[id] = 1;
        else
            NextState[id] = CurrentState[id];
    }
}