#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        // Use ternary in multiplication to simplify condition
        ingrad[tid] = (indata[tid] > 0) ? outgrad[tid] : 0;
    }
}