#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_one(float * prp_0,int sz)
{
    // Calculate flattened global thread index
    int index = blockIdx.z * blockDim.z * sz * sz + blockIdx.y * blockDim.y * sz + blockIdx.x * blockDim.x 
                + threadIdx.z * sz * sz + threadIdx.y * sz + threadIdx.x;

    if(index < sz * sz * sz) // Boundary check to prevent out-of-bounds access
    {
        prp_0[index] = 1.0f;
    }
}