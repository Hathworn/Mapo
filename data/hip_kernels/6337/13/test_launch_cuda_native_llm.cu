#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void test_launch_cuda_native(float * scalar, float * vector, int sxy, int sx , int sy , int sz , int stride)
{
    int id[3] = {
        threadIdx.x + blockIdx.x * blockDim.x,
        threadIdx.y + blockIdx.y * blockDim.y,
        threadIdx.z + blockIdx.z * blockDim.z
    };

    // Exit early if any index is out of bounds
    if (id[0] >= sx || id[1] >= sy || id[2] >= sz) { return; }

    int index = id[2] * sxy + id[1] * sx + id[0];
    float s = scalar[index];

    // Precompute vector base index for efficiency
    int vector_index_base = index;
    
    float v[3] = {
        vector[vector_index_base + 0 * stride],
        vector[vector_index_base + 1 * stride],
        vector[vector_index_base + 2 * stride]
    };

    printf("Grid point from CUDA %d %d %d     scalar: %f  vector: %f %f %f \n", id[0], id[1], id[2], s, v[0], v[1], v[2]);
}