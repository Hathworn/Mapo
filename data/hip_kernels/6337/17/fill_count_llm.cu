#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_count(float * prp_0, int sz)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;

    // Optimize array access and update when within bounds
    int index = idz * sz * sz + idy * sz + idx;
    if (idx < sz && idy < sz && idz < sz) {
        prp_0[index] = index; // Simplified computation and ensure in-bounds operation
    }
}