#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Description:
*/

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int d4, int scale_factor_t, int scale_factor_y, int scale_factor_x, int off_time, int off_x, int off_y)
{
    int x, y, t, z, w;
    w = ii % d4;
    ii = ii / d4;
    z = ii % d3;
    ii = ii / d3;
    t = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    t = t * scale_factor_t + off_time;
    w = w * scale_factor_x + off_x;
    z = z * scale_factor_y + off_y;
    d2 *= scale_factor_t;
    d3 *= scale_factor_y;
    d4 *= scale_factor_x;
    return (((((x * d1 + y) * d2) + t) * d3) + z) * d4 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int d4, int scale_factor_t, int scale_factor_y, int scale_factor_x)
{
    int x, y, t, z, w;
    w = ii % d4;
    ii = ii / d4;
    z = ii % d3;
    ii = ii / d3;
    t = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor_x;
    z = z / scale_factor_y;
    t = t / scale_factor_t;
    d2 /= scale_factor_t;
    d3 /= scale_factor_y;
    d4 /= scale_factor_x;
    return (((((x * d1 + y) * d2) + t) * d3) + z) * d4 + w;
}

__global__ void downscale(float *gradInput_data, float *gradOutput_data, long no_elements, int scale_factor_t, int scale_factor_y, int scale_factor_x, int d1, int d2, int d3, int d4)
{
    long ii = threadIdx.x + blockDim.x * blockIdx.x + threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
    if (ii >= no_elements) return;

    // Unrolling the loops to maximize performance
    int scale_factor_t_2 = scale_factor_t / 2;
    int scale_factor_y_2 = scale_factor_y / 2;
    int scale_factor_x_2 = scale_factor_x / 2;

    for (int i = 0; i < scale_factor_t_2; i++){
#pragma unroll
        for(int j = 0; j < scale_factor_y_2; j++){
#pragma unroll
            for(int k = 0; k < scale_factor_x_2; k++){
                int ipidx1 = translate_idx_inv(ii, d1, d2, d3, d4, scale_factor_t, scale_factor_y, scale_factor_x, 2*i, 2*j, 2*k);
                int ipidx2 = translate_idx_inv(ii, d1, d2, d3, d4, scale_factor_t, scale_factor_y, scale_factor_x, 2*i+1, 2*j, 2*k);
                int ipidx3 = translate_idx_inv(ii, d1, d2, d3, d4, scale_factor_t, scale_factor_y, scale_factor_x, 2*i, 2*j+1, 2*k);
                int ipidx4 = translate_idx_inv(ii, d1, d2, d3, d4, scale_factor_t, scale_factor_y, scale_factor_x, 2*i+1, 2*j+1, 2*k);
                int ipidx5 = translate_idx_inv(ii, d1, d2, d3, d4, scale_factor_t, scale_factor_y, scale_factor_x, 2*i, 2*j, 2*k+1);
                int ipidx6 = translate_idx_inv(ii, d1, d2, d3, d4, scale_factor_t, scale_factor_y, scale_factor_x, 2*i+1, 2*j, 2*k+1);
                int ipidx7 = translate_idx_inv(ii, d1, d2, d3, d4, scale_factor_t, scale_factor_y, scale_factor_x, 2*i, 2*j+1, 2*k+1);
                int ipidx8 = translate_idx_inv(ii, d1, d2, d3, d4, scale_factor_t, scale_factor_y, scale_factor_x, 2*i+1, 2*j+1, 2*k+1);

                gradInput_data[ii] += gradOutput_data[ipidx1] + gradOutput_data[ipidx2] + gradOutput_data[ipidx3] + gradOutput_data[ipidx4] +
                                      gradOutput_data[ipidx5] + gradOutput_data[ipidx6] + gradOutput_data[ipidx7] + gradOutput_data[ipidx8];
            }
        }
    }
}