#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Improved index calculation to reduce arithmetic inside the loop
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Unrolling the loop to improve performance by doing more work per iteration
    for (unsigned int i = idx; i < numEls; i += stride * 4) {
        if (i < numEls) dest[i] = a[i] + b[i];
        if (i + stride < numEls) dest[i + stride] = a[i + stride] + b[i + stride];
        if (i + 2 * stride < numEls) dest[i + 2 * stride] = a[i + 2 * stride] + b[i + 2 * stride];
        if (i + 3 * stride < numEls) dest[i + 3 * stride] = a[i + 3 * stride] + b[i + 3 * stride];
    }
}