#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void polynomial_expansion(float* poly, int degree, int n, float* array) {
    // Calculate index using block and thread identifiers
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < n) {
        float out = 0.0;
        float xtothepowerof = 1.0;

        // Loop to compute polynomial expansion
        for (int x = 0; x <= degree; ++x) {
            out += xtothepowerof * poly[x];
            
            // Inline multiplication to reduce redundancy
            xtothepowerof *= array[index];
        }

        // Store result into the array
        array[index] = out;
    }
}