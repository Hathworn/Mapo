#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use blockIdx.x as an index to allow kernel execution on multiple elements
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index == 0) {
        // Only the first thread performs the division to prevent race conditions
        *accuracy /= N;
    }
}