#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

__global__ void vectorAdditionCUDA(const float* a, const float* b, float* c, int n)
{
    int ii = blockDim.x * blockIdx.x + threadIdx.x;
    // Use if condition in a non-divergent manner
    if (ii < n) {
        c[ii] = a[ii] + b[ii];
    }
}