#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelA(float* r, float* x, float* y, float* z, int size)
{
    // Calculate global index for current thread
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Use global index to iterate over the data with block-wide stride
    for (int i = index; i < size; i += blockDim.x * gridDim.x) {
        r[i] = x[i] * y[i] + z[i];
    }
}