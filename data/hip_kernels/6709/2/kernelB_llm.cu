#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop over each global thread, with stride of total threads
    for (; i < size; i += blockDim.x * gridDim.x) {
        r[i] = x[i] * y[i] + z[i];
    }
}