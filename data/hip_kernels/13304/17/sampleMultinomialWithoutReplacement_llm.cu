#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;
        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }
    if (start == size) {
        start = 0;
    }
    return start;
}

__global__ void sampleMultinomialWithoutReplacement(hiprandStateMtgp32* state, int totalSamples, int sample, float* dest, long distributions, int categories, float* origDist, float* normDistPrefixSum) {
    // Efficiently compute the curDist for this thread
    long curDistBase = blockIdx.x * blockDim.y + threadIdx.y;
    
    if (curDistBase < distributions) {
        float r = hiprand_uniform(&state[blockIdx.x]);
        if (threadIdx.x == 0) {
            // Perform binary search to find the choice
            int choice = binarySearchForMultinomial(
                normDistPrefixSum + curDistBase * categories,
                categories,
                r);

            // Update the result array and original distribution
            dest[curDistBase * totalSamples + sample] = static_cast<float>(choice + 1);
            origDist[curDistBase * categories + choice] = 0.0f;
        }
    }
}
```
