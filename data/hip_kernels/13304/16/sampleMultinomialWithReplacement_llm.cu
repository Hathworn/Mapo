#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;

        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        // No probability mass or precision problems; just return the
        // first element
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithReplacement(hiprandStateMtgp32* state, int totalSamples, float* dest, long distributions, int categories, float* normDistPrefixSum) {
    // Optimize by using blockDim.y as part of loop increment
    for (long curDist = blockIdx.x; curDist < distributions; curDist += gridDim.x) {
        for (int sampleBase = threadIdx.y; sampleBase < totalSamples; sampleBase += blockDim.y) {
            // Each thread handles its own sample
            float r = hiprand_uniform(&state[blockIdx.x]);

            // Avoid divergence by using the condition within each sample loop
            if (sampleBase < totalSamples) {
                // Find the bucket that a uniform sample lies in
                int choice = binarySearchForMultinomial(
                    normDistPrefixSum + curDist * categories,
                    categories,
                    r);

                // Torch indices are 1-based
                dest[curDist * totalSamples + sampleBase] = (float) choice + 1.0f;
            }
        }
    }
}