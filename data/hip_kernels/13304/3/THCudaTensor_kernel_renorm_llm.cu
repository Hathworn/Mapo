#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const long size, const float maxnorm)
{
    __shared__ float buffer[32];
    long tx = threadIdx.x;
    long bx = blockIdx.x;
    long step = blockDim.x;

    // Use a pointer to the current row
    float *row = data + size * bx;
    buffer[tx] = 0.0f;

    // Calculate local sums in parallel
    for (long i = tx; i < size; i += step)
    {
        buffer[tx] += powf(fabsf(row[i]), value);
    }
    
    // Perform parallel reduction within block
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        __syncthreads(); // Ensure all threads have written their sums
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
    }
    
    // Normalize only if the first thread
    if (tx == 0)
    {
        float norm = powf(buffer[0], 1.0f / value);
        if (norm > maxnorm)
        {
            norm = maxnorm / (norm + 1e-7f);
            buffer[0] = norm; // Store normalization factor for reuse
        }
    }

    __syncthreads(); // Ensure normalization factor is computed

    // Renormalize if needed
    if (buffer[0] < 1.0f) // Check if normalization factor is computed
    {
        for (long i = tx; i < size; i += step)
        {
            row[i] *= buffer[0];
        }
    }
}