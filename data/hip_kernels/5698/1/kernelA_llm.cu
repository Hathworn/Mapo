#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelA(float* r, float* x, float* y, float* z, int size)
{
    // Calculate the global index for the current thread.
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single loop with index stride by total grid size.
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < size; i += stride) {
        r[i] = x[i] * y[i] + z[i];
    }
}