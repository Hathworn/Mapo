#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

#define GRID_SIZE 32
#define SHARED_MEM 16384

__global__ void findY(float *x, float *y, int n, float h, float z, int zLoc, float *returnVal) {
    // Use an atomic addition for thread-safe accumulation
    __shared__ float sum;
    if (threadIdx.x == 0) sum = 0;  // Initialize shared memory once per block
    __syncthreads();

    float localSum = 0;
    int localCount = 0;
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        if (abs(x[i] - z) < h) {
            localSum += y[i];
            localCount++;
        }
    }

    atomicAdd(&sum, localSum);  // Accumulate results atomically
    __syncthreads();

    if (threadIdx.x == 0) {
        *returnVal = sum / max(localCount, 1);  // Avoid division by zero
    }
}