#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shift_cypher(unsigned int *input_array, unsigned int *output_array, unsigned int shift_amount, unsigned int alphabet_max, unsigned int array_length)
{
    // Define thread index for 1D grid
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Bounds check to ensure valid access
    if (idx < array_length)
    {
        // Perform shift and modulo operation for cypher
        output_array[idx] = (input_array[idx] + shift_amount) % alphabet_max;
    }
}