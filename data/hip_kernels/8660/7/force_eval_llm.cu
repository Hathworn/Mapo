#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void force_eval(float4 *set_A, float4 *set_B, int *indices, float4 *force_vectors, int array_length)
{
    // Compute unique global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds to prevent accessing out of array
    if (idx < array_length) {
        // Load indices from global to register
        int indexA = indices[idx * 2];
        int indexB = indices[idx * 2 + 1];

        // Load elements from global memory to registers
        float4 a = set_A[indexA];
        float4 b = set_B[indexB];

        // Calculate the force vector (example operation)
        float4 force;
        force.x = a.x - b.x;
        force.y = a.y - b.y;
        force.z = a.z - b.z;
        force.w = a.w - b.w;

        // Store the result back to global memory
        force_vectors[idx] = force;
    }
}