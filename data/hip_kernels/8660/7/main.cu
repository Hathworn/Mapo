#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "force_eval.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float4 *set_A = NULL;
hipMalloc(&set_A, XSIZE*YSIZE);
float4 *set_B = NULL;
hipMalloc(&set_B, XSIZE*YSIZE);
int *indices = NULL;
hipMalloc(&indices, XSIZE*YSIZE);
float4 *force_vectors = NULL;
hipMalloc(&force_vectors, XSIZE*YSIZE);
int array_length = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
force_eval<<<gridBlock, threadBlock>>>(set_A,set_B,indices,force_vectors,array_length);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
force_eval<<<gridBlock, threadBlock>>>(set_A,set_B,indices,force_vectors,array_length);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
force_eval<<<gridBlock, threadBlock>>>(set_A,set_B,indices,force_vectors,array_length);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}