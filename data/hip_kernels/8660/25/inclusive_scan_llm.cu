#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized inclusive_scan kernel function
__global__ void inclusive_scan(const unsigned int *input, unsigned int *result)
{
    extern __shared__ unsigned int sdata[];

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input into shared memory
    sdata[threadIdx.x] = input[i];
    __syncthreads();

    // Hillis-Steele scan pattern
    for (int offset = 1; offset < blockDim.x; offset <<= 1)
    {
        unsigned int temp = 0;
        if(threadIdx.x >= offset)
        {
            temp = sdata[threadIdx.x - offset];
        }
        __syncthreads();

        // Sum two elements conditionally
        sdata[threadIdx.x] += temp;
        __syncthreads();
    }

    // Write results back
    result[i] = sdata[threadIdx.x];
}