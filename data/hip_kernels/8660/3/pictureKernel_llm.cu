#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pictureKernel(float* d_pix, int X, int Y) {
    // Calculate global linear thread index
    int idx = blockIdx.y * blockDim.y * X + blockIdx.x * blockDim.x + threadIdx.y * X + threadIdx.x;

    // Ensure thread is within bounds before processing
    if (threadIdx.x + blockIdx.x * blockDim.x < X && threadIdx.y + blockIdx.y * blockDim.y < Y) {
        d_pix[idx] *= 2; // Efficient memory access with linear indexing
    }
}