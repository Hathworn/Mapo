#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmulKernel(float* mat1, float* mat2, float* matP, int dim) {
    // Use shared memory for better performance
    extern __shared__ float shared[];
    float* shared_mat1 = shared;
    float* shared_mat2 = shared + blockDim.y * dim;
    
    int thread_x = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_y = blockIdx.y * blockDim.y + threadIdx.y;

    float P_value = 0.0f;
    
    for (int tile = 0; tile < dim; tile += blockDim.x) {
        // Load tiles into shared memory
        shared_mat1[thread_y * dim + threadIdx.x] = mat1[thread_y * dim + tile + threadIdx.x];
        shared_mat2[threadIdx.y * dim + thread_x] = mat2[(tile + threadIdx.y) * dim + thread_x];
        
        __syncthreads(); // Synchronize to ensure tiles are loaded
        
        // Perform partial dot product using tiles
        for (int i = 0; i < blockDim.x; ++i) {
            P_value += shared_mat1[thread_y * dim + i] * shared_mat2[i * dim + thread_x];
        }
        
        __syncthreads(); // Synchronize before loading next tiles
    }

    if (thread_x < dim && thread_y < dim) {
        matP[thread_y * dim + thread_x] = P_value;
    }
}