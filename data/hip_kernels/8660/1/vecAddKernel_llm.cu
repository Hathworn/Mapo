#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
void vectorAdd(double* A, double* B, double* C,int n);

__global__ void vecAddKernel(double* A, double* B, double* C, int n) {
    // Calculate global index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Check index bounds and perform vector addition
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}