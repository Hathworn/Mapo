#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adjacent_difference_simple(int *result, int *input)
{
    // Compute this thread's global index
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory to reduce global memory access
    __shared__ int shared_input[blockDim.x];
    if (threadIdx.x < blockDim.x)
    {
        shared_input[threadIdx.x] = input[i];
    }
    __syncthreads();

    // Compute difference only if i > 0
    if (i > 0)
    {
        // Use shared memory to fetch values
        int x_i = shared_input[threadIdx.x];
        int x_i_minus_one = (threadIdx.x == 0) ? input[i - 1] : shared_input[threadIdx.x - 1];

        // Compute the difference using values stored in registers
        result[i] = x_i - x_i_minus_one;
    }
}