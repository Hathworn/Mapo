#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forwardDifference2DKernel(const int cols, const int rows, const float* data, float* dx, float* dy) {
    // Calculate global thread indices
    int idy = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;

    // Ensure thread indices within bounds
    if (idy < cols - 1 && idx < rows - 1) {
        // Compute linear index
        const auto index = idx + rows * idy;
        
        // Calculate forward differences
        dx[index] = data[index + 1] - data[index];
        dy[index] = data[index + rows] - data[index];
    }
}