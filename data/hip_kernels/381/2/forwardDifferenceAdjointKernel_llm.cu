#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forwardDifferenceAdjointKernel(const int len, const float* source, float* target) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
    // Unroll loop for performance
    while (idx < len - 1) {
        target[idx] = -source[idx] + source[idx - 1];
        idx += blockDim.x * gridDim.x;
    }
}