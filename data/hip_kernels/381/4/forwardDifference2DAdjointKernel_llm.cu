#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forwardDifference2DAdjointKernel(const int cols, const int rows, const float* dx, const float* dy, float* target) {
    // Calculate the globally unique ID for the thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int idy = blockIdx.y * blockDim.y + threadIdx.y + 1;
    
    // Calculate stride for grid
    int strideX = blockDim.x * gridDim.x;
    int strideY = blockDim.y * gridDim.y;

    // Loop through all elements assigned to this thread
    for (int y = idy; y < cols - 1; y += strideY) {
        for (int x = idx; x < rows - 1; x += strideX) {
            const int index = x + rows * y;
            // Compute the forward difference
            target[index] = -dx[index] + dx[index - 1] - dy[index] + dy[index - rows];
        }
    }
}