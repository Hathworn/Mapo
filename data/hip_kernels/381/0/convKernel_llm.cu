#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convKernel(const float* __restrict__ source, const float* __restrict__ kernel, float* __restrict__ target, const int len) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if out-of-bounds
    if (idx >= len) return;

    float value = 0.0f;

    // Loop unrolling assuming len is a multiple of 4 can improve performance
    for (int i = 0; i < len; i += 4) {
        // Use texture memory to potentially increase memory throughput
        value += source[i] *
                 kernel[(len + len / 2 + idx - i) % len];
        value += source[i+1] *
                 kernel[(len + len / 2 + idx - i-1) % len];
        value += source[i+2] *
                 kernel[(len + len / 2 + idx - i-2) % len];
        value += source[i+3] *
                 kernel[(len + len / 2 + idx - i-3) % len];
    }

    // Write the result
    target[idx] = value;
}