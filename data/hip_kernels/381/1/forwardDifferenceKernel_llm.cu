#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forwardDifferenceKernel(const int len, const float* source, float* target) {
    // Calculate unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
    
    // Ensure valid index within data bounds
    if (idx < len - 1) {
        // Perform computation
        target[idx] = source[idx + 1] - source[idx];
    }
}