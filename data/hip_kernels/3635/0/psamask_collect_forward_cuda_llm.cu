#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA: grid stride looping
#ifndef CUDA_KERNEL_LOOP
#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)
#endif

__global__ void psamask_collect_forward_cuda(const int nthreads, const int feature_H_, const int feature_W_, const int mask_H_, const int mask_W_, const int half_mask_H_, const int half_mask_W_, const float* mask_data, float* buffer_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        const int w = index % feature_W_;
        const int h = (index / feature_W_) % feature_H_;
        const int n = index / feature_W_ / feature_H_;
        
        // Effective mask region : [hstart, hend) x [wstart, wend) with mask-indexed
        const int hstart = max(0, half_mask_H_ - h);
        const int hend = min(mask_H_, feature_H_ + half_mask_H_ - h);
        const int wstart = max(0, half_mask_W_ - w);
        const int wend = min(mask_W_, feature_W_ + half_mask_W_ - w);
        
        // Precompute index offsets for efficiency
        const int feature_offset = n * feature_H_ * feature_W_;
        const int mask_offset = n * mask_H_ * mask_W_;
        
        for (int hidx = hstart; hidx < hend; hidx++) {
            for (int widx = wstart; widx < wend; widx++) {
                const int h_feature_idx = hidx + h - half_mask_H_;
                const int w_feature_idx = widx + w - half_mask_W_;
                buffer_data[(feature_offset + h_feature_idx * feature_W_ + w_feature_idx) * feature_H_ * feature_W_ + h * feature_W_ + w] =
                    mask_data[((mask_offset + hidx * mask_W_ + widx) * feature_H_ + h) * feature_W_ + w];
            }
        }
    }
}