#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA: grid stride looping
#ifndef CUDA_KERNEL_LOOP
#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)
#endif

__global__ void psamask_distribute_backward_cuda(const int nthreads, const int feature_H_, const int feature_W_, const int mask_H_, const int mask_W_, const int half_mask_H_, const int half_mask_W_, const float* buffer_diff, float* mask_diff) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        const int w = index % feature_W_;
        const int h = (index / feature_W_) % feature_H_;
        const int n = index / feature_W_ / feature_H_;
        // Precompute expressions used in the loops to avoid recalculations.
        int indexOffset = (n * feature_H_ * feature_W_ + h * feature_W_ + w) * feature_H_ * feature_W_ - half_mask_H_ * feature_W_ - half_mask_W_;
        const int hstart = max(0, half_mask_H_ - h);
        const int hend = min(mask_H_, feature_H_ + half_mask_H_ - h);
        const int wstart = max(0, half_mask_W_ - w);
        const int wend = min(mask_W_, feature_W_ + half_mask_W_ - w);

        for (int hidx = hstart; hidx < hend; ++hidx) {
            for (int widx = wstart; widx < wend; ++widx) {
                mask_diff[((n * mask_H_ * mask_W_ + hidx * mask_W_ + widx) * feature_H_ + h) * feature_W_ + w] =
                    buffer_diff[indexOffset + (hidx + h) * feature_W_ + (widx + w)];
            }
        }
    }
}