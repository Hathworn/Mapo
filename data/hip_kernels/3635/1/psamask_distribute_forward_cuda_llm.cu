#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA: grid stride looping
#ifndef CUDA_KERNEL_LOOP
#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)
#endif

__global__ void psamask_distribute_forward_cuda(const int nthreads, const int feature_H_, const int feature_W_, const int mask_H_, const int mask_W_, const int half_mask_H_, const int half_mask_W_, const float* mask_data, float* buffer_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
        const int w = index % feature_W_;
        const int h = (index / feature_W_) % feature_H_;
        const int n = index / feature_W_ / feature_H_;
        const int hstart = max(0, half_mask_H_ - h);
        const int hend = min(mask_H_, feature_H_ + half_mask_H_ - h);
        const int wstart = max(0, half_mask_W_ - w);
        const int wend = min(mask_W_, feature_W_ + half_mask_W_ - w);

        const int buffer_offset = n * feature_H_ * feature_W_ * feature_H_ * feature_W_ + h * feature_W_ + w;
        const int mask_base_index = n * mask_H_ * mask_W_ * feature_H_ * feature_W_ + h * feature_W_ + w;

        for (int hidx = hstart; hidx < hend; hidx++) {
            for (int widx = wstart; widx < wend; widx++) {
                // Compute flat indices for linear indexing
                int buffer_idx = buffer_offset + (hidx + h - half_mask_H_) * feature_W_ + (widx + w - half_mask_W_);
                int mask_idx = mask_base_index + hidx * mask_W_ * feature_H_ * feature_W_ + widx * feature_H_ * feature_W_;
                buffer_data[buffer_idx] = mask_data[mask_idx];
            }
        }
    }
}