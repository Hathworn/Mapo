#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void countQueens(int* frontQueensPos, int* data, int* numFQP)
{
    int localResult = 0;
    int thisThread = blockIdx.z * (blockDim.x * blockDim.y * gridDim.x * gridDim.y) + 
                     blockIdx.y * (blockDim.x * blockDim.y * gridDim.x) + 
                     blockIdx.x * (blockDim.x * blockDim.y) + 
                     threadIdx.y * blockDim.x + threadIdx.x;

    // Boundary check for halting threads outside the work scope early
    if (blockIdx.x >= QUEENS || blockIdx.y >= QUEENS || threadIdx.x >= QUEENS || threadIdx.y >= QUEENS)
        return;

    // Allocate queen positions in shared memory to reduce frequent global memory access
    __shared__ int sharedQueenPos[QUEENS];
    sharedQueenPos[3] = blockIdx.x;
    sharedQueenPos[4] = blockIdx.y;
    sharedQueenPos[5] = threadIdx.x;
    sharedQueenPos[6] = threadIdx.y;

    // Unrolling the 'for' loop for reduced branch divergence
    if ((sharedQueenPos[4] - 4) == (sharedQueenPos[3] - 3) || (sharedQueenPos[4] + 4) == (sharedQueenPos[3] + 3) || sharedQueenPos[4] == sharedQueenPos[3]) return;
    if ((sharedQueenPos[5] - 5) == (sharedQueenPos[3] - 3) || (sharedQueenPos[5] + 5) == (sharedQueenPos[3] + 3) || sharedQueenPos[5] == sharedQueenPos[3] || (sharedQueenPos[5] - 5) == (sharedQueenPos[4] - 4) || (sharedQueenPos[5] + 5) == (sharedQueenPos[4] + 4) || sharedQueenPos[5] == sharedQueenPos[4]) return;
    if ((sharedQueenPos[6] - 6) == (sharedQueenPos[3] - 3) || (sharedQueenPos[6] + 6) == (sharedQueenPos[3] + 3) || sharedQueenPos[6] == sharedQueenPos[3] || (sharedQueenPos[6] - 6) == (sharedQueenPos[4] - 4) || (sharedQueenPos[6] + 6) == (sharedQueenPos[4] + 4) || sharedQueenPos[6] == sharedQueenPos[4] ||
        (sharedQueenPos[6] - 6) == (sharedQueenPos[5] - 5) || (sharedQueenPos[6] + 6) == (sharedQueenPos[5] + 5) || sharedQueenPos[6] == sharedQueenPos[5]) return;

    int totalFQP = numFQP[0] / 3;

    for (int FQP_number = 0; FQP_number < totalFQP; FQP_number++) {
        for (int i = 0; i < 3; i++)
            sharedQueenPos[i] = frontQueensPos[(FQP_number * 3) + i];

        bool legal = true;

        for (int i = 3; i <= 6; i++) {
            for (int j = 0; j < 3; j++) {
                if ((sharedQueenPos[i] - i) == (sharedQueenPos[j] - j) || (sharedQueenPos[i] + i) == (sharedQueenPos[j] + j) || sharedQueenPos[i] == sharedQueenPos[j]) {
                    legal = false;
                    break;
                }
            }
            if (!legal)
                break;
        }
        if (!legal)
            continue;

        int posNow = 7;
        sharedQueenPos[posNow] = -1;
        while (posNow > 6) {
            sharedQueenPos[posNow]++;
            while (sharedQueenPos[posNow] < QUEENS) {
                legal = true;
                for (int j = posNow - 1; j >= 0; j--) {
                    if ((sharedQueenPos[posNow] - posNow) == (sharedQueenPos[j] - j) || (sharedQueenPos[posNow] + posNow) == (sharedQueenPos[j] + j) || sharedQueenPos[posNow] == sharedQueenPos[j]) {
                        legal = false;
                        break;
                    }
                }
                if (!legal)
                    sharedQueenPos[posNow]++;
                else
                    break;
            }
            if (sharedQueenPos[posNow] < QUEENS) {
                if (posNow == (QUEENS - 1)) {
                    localResult++;
                    posNow--;
                }
                else {
                    posNow++;
                    sharedQueenPos[posNow] = -1;
                }
            }
            else
                posNow--;
        }
    }
    data[thisThread] = localResult;
}