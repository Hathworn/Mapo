#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void countQueens(int* frontQueensPos, int* data, int* numFQP)
{
    int localResult = 0;
    int thisThread = ((blockIdx.x * gridDim.x + blockIdx.y) * gridDim.y + threadIdx.x) * blockDim.x + threadIdx.y;

    if (blockIdx.x >= QUEENS || blockIdx.y >= QUEENS || threadIdx.x >= QUEENS || threadIdx.y >= QUEENS)
        return;

    int queenPos[QUEENS];  // Use stack allocation instead of new

    queenPos[3] = blockIdx.x;
    queenPos[4] = blockIdx.y;
    queenPos[5] = threadIdx.x;
    queenPos[6] = threadIdx.y;

    // Initial conflict checks
    for (int i = 4; i <= 6; i++) {
        for (int j = 3; j < i; j++) {
            if ((queenPos[i] - i) == (queenPos[j] - j) || (queenPos[i] + i) == (queenPos[j] + j) || queenPos[i] == queenPos[j]) {
                return;
            }
        }
    }

    int totalFQP = numFQP[0] / 3;

    for (int FQP_number = 0; FQP_number < totalFQP; FQP_number++) {
        for (int i = 0; i < 3; i++)
            queenPos[i] = frontQueensPos[(FQP_number * 3) + i];

        bool legal = true;
        
        // Additional conflict checks
        for (int i = 3; i <= 6; i++) {
            for (int j = 0; j < 3; j++) {
                if ((queenPos[i] - i) == (queenPos[j] - j) || (queenPos[i] + i) == (queenPos[j] + j) || queenPos[i] == queenPos[j]) {
                    legal = false;
                    break;
                }
            }
            if (!legal) break;
        }
        if (!legal) continue;

        int posNow = 7;
        queenPos[posNow] = -1;
        while (posNow > 6) {
            queenPos[posNow]++;
            while (queenPos[posNow] < QUEENS) {
                legal = true;
                for (int j = posNow - 1; j >= 0; j--) {
                    if ((queenPos[posNow] - posNow) == (queenPos[j] - j) || 
                        (queenPos[posNow] + posNow) == (queenPos[j] + j) || 
                        queenPos[posNow] == queenPos[j]) {
                        legal = false;
                        break;
                    }
                }
                if (!legal)
                    queenPos[posNow]++;
                else
                    break;
            }
            if (queenPos[posNow] < QUEENS) {
                if (posNow == (QUEENS - 1)) {
                    localResult++;
                    posNow--;
                } else {
                    posNow++;
                    queenPos[posNow] = -1;
                }
            } else {
                posNow--;
            }
        }
    }
    data[thisThread] = localResult;
}