#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// tatami.cu

const unsigned nMax(100000000);
const unsigned nMaxSqrt(sqrt(nMax));

__global__ void even(unsigned* v, unsigned base)
{
    unsigned i = (blockIdx.x * blockDim.x + threadIdx.x + base) * 2 + 8;
    unsigned k2 = i + 3;
    unsigned k3 = i + i - 4;
    unsigned k4;

    // Use shared memory for synchronization as less as possible
    while ((k2 <= k3) && ((i * k2) < nMax))
    {
        // Precompute constant table value to reduce atomic operations
        k4 = min((nMax - 1) / i, k3);
        for (unsigned j = k2; j <= k4; ++j)
        {
            // Atomic increment to prevent race condition
            atomicInc(&v[i * j / 2], 0xffffffff);
        }
        k2 += i + 1;
        k3 += i - 1;
        __syncthreads(); // Only necessary here to sync on next iteration
    }
}