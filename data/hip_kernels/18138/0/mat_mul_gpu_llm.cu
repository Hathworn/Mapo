#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mat_mul_gpu(float* vec_one, float* vec_two, float* ret_vec, int vec_one_row, int vec_one_col, int vec_two_col) {
    // Compute global thread coordinates
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Linearize coordinates for data access
    if ((row < vec_one_row) && (col < vec_two_col)) {
        float cum_sum = 0.0;

        // Unroll loop for better performance
        #pragma unroll
        for (int k = 0; k < vec_one_col; k++) {
            cum_sum += vec_one[row * vec_one_col + k] * vec_two[k * vec_two_col + col];
        }
        ret_vec[row * vec_two_col + col] = cum_sum;
    }
}