#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixAddKernel1(float* ans, float* M, float* N, int size) {
    // Use shared memory to optimize memory access
    __shared__ float sharedM[blockDim.y][blockDim.x];
    __shared__ float sharedN[blockDim.y][blockDim.x];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (row < size && col < size) {
        sharedM[threadIdx.y][threadIdx.x] = M[row * size + col];
        sharedN[threadIdx.y][threadIdx.x] = N[row * size + col];
    }

    __syncthreads();  // Ensure shared memory is loaded

    // Perform addition using shared memory
    if (row < size && col < size) {
        ans[row * size + col] = sharedM[threadIdx.y][threadIdx.x] + sharedN[threadIdx.y][threadIdx.x];
    }
}