#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif

// generate a random square matrix
__global__ void matMulKernel25(float* P, float* M, float* N, int width) {
    __shared__ float Mds25[25][25];
    __shared__ float Nds25[25][25];
    int bx = blockIdx.x; 
    int by = blockIdx.y;
    int tx = threadIdx.x; 
    int ty = threadIdx.y;
    int row = by * 25 + ty; 
    int col = bx * 25 + tx;
    float pVal = 0;

    for (int ph = 0; ph < width / 25; ++ph) {
        // Load M and N into shared memory with coalesced accesses
        Mds25[ty][tx] = M[row * width + ph * 25 + tx];
        Nds25[ty][tx] = N[(ph * 25 + ty) * width + col];
        __syncthreads();

        // Reduce number of synchronizations, loop unrolling
        #pragma unroll
        for (int k = 0; k < 25; ++k) {
            pVal += Mds25[ty][k] * Nds25[k][tx];
        }
        __syncthreads(); // Needed to ensure data integrity for next iteration
    }
    
    P[row * width + col] = pVal;
}