#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixAddKernel2(float* ans, float* M, float* N, int size) {
    // Use both row and column indices to fully utilize thread parallelism
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure both row and col are within bounds
    if(row < size && col < size) {
        int index = row * size + col;
        ans[index] = M[index] + N[index];
    }
}