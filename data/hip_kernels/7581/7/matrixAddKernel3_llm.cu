#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixAddKernel3(float* ans, float* M, float* N, int size) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = row * size + col;

    // Ensure both row and col are within bounds
    if (row < size && col < size) {
        ans[idx] = M[idx] + N[idx];
    }
}