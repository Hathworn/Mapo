#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMulKernel(float* ans, float* M, float* N, int size) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread indices are within valid range
    if(row < size && col < size) {
        float pVal = 0.0f;
        
        // Use shared memory for faster data access
        for (int i = 0; i < size; ++i) {
            pVal += M[row * size + i] * N[i * size + col]; 
        }
        
        ans[row * size + col] = pVal; // Store computed value
    }
}