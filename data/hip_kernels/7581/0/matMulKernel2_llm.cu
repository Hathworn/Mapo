#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif

// generate a random square matrix
__global__ void matMulKernel2(float* P, float* M, float* N, int width) {
    __shared__ float Mds[2][2];
    __shared__ float Nds[2][2];
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int row = by * 2 + ty, col = bx * 2 + tx;
    float pVal = 0;

    for (int ph = 0; ph < width / 2; ++ph) {
        // Load data into shared memory
        Mds[ty][tx] = M[row * width + ph * 2 + tx];
        Nds[ty][tx] = N[(ph * 2 + ty) * width + col];
        __syncthreads(); // Sync threads to make sure all data is loaded

        // Compute partial product contributions
        #pragma unroll // Unroll loop for better performance
        for (int k = 0; k < 2; ++k)
            pVal += Mds[ty][k] * Nds[k][tx];
        __syncthreads(); // Ensure all threads complete computation
    }

    // Write final result to global memory
    P[row * width + col] = pVal;
}