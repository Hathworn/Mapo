#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif

// Optimized global kernel function matMulKernel10
__global__ void matMulKernel10(float* P, float* M, float* N, int width) {
    __shared__ float Mds10[10][10];
    __shared__ float Nds10[10][10];
    
    int bx = blockIdx.x; 
    int by = blockIdx.y;
    int tx = threadIdx.x; 
    int ty = threadIdx.y;
    
    int row = by * 10 + ty; 
    int col = bx * 10 + tx;
    float pVal = 0;

    for (int ph = 0; ph < width / 10; ++ph) {
        // Load a tile of M into shared memory
        Mds10[ty][tx] = M[row * width + ph * 10 + tx];
        // Load a tile of N into shared memory
        Nds10[ty][tx] = N[(ph * 10 + ty) * width + col];
        __syncthreads();

        #pragma unroll  // Unroll the loop for better performance
        for (int k = 0; k < 10; ++k) {
            pVal += Mds10[ty][k] * Nds10[k][tx];
        }
        __syncthreads();
    }
    // Write the result to the output matrix
    P[row * width + col] = pVal;
}