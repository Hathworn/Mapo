#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif

// Optimized matrix multiplication kernel
__global__ void matMulKernel4(float* P, float* M, float* N, int width) {
    __shared__ float Mds4[4][4];
    __shared__ float Nds4[4][4];
    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    int row = by * 4 + ty; int col = bx * 4 + tx;
    float pVal = 0.0f;  // Use type-specific literal

    // Use unrolling to reduce loop overhead
    for (int ph = 0; ph < width / 4; ++ph) {
        Mds4[ty][tx] = M[row * width + ph * 4 + tx];
        Nds4[ty][tx] = N[(ph * 4 + ty) * width + col];
        __syncthreads();

        // Unroll inner loop
        #pragma unroll
        for (int k = 0; k < 4; ++k) {
            pVal += Mds4[ty][k] * Nds4[k][tx];
        }
        __syncthreads();
    }
    P[row * width + col] = pVal;
}