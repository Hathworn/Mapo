#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nms_kernel(const int num_per_thread, const float threshold, const int num_detections, const int *indices, float *scores, const float *classes, const float4 *boxes) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Compute global thread index
    if (i >= num_detections * num_per_thread) return;  // Exit if thread index out of bounds

    int detection_index = i / num_per_thread;  // Determine which detection to process
    int local_index = i % num_per_thread;      // Determine which element to process within this detection

    for (int m = 0; m < num_detections; m++) {
        if (detection_index < num_detections && m < detection_index && scores[m] > 0.0f) {
            int idx = indices[detection_index];
            int max_idx = indices[m];
            int icls = classes[idx];
            int mcls = classes[max_idx];
            
            if (mcls == icls) {
                float4 ibox = boxes[idx];
                float4 mbox = boxes[max_idx];
                float x1 = max(ibox.x, mbox.x);
                float y1 = max(ibox.y, mbox.y);
                float x2 = min(ibox.z, mbox.z);
                float y2 = min(ibox.w, mbox.w);
                float w = max(0.0f, x2 - x1 + 1);
                float h = max(0.0f, y2 - y1 + 1);
                float iarea = (ibox.z - ibox.x + 1) * (ibox.w - ibox.y + 1);
                float marea = (mbox.z - mbox.x + 1) * (mbox.w - mbox.y + 1);
                float inter = w * h;
                float overlap = inter / (iarea + marea - inter);
                
                if (overlap > threshold) {
                    scores[detection_index] = 0.0f;
                }
            }
        }
        __syncthreads();  // Sync all threads to ensure consistent updates
    }
}