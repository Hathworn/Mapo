#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define WARP_SIZE 32 // # of threads that are executed together (constant valid on most hardware)

// CPU entry point for kernel to add "scalar" to every element of the input array
__global__ void _cuda_add_scalar(int *in, int scalar, int n)
{
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride only once outside loop for efficiency
    for (; globalIdx < n; globalIdx += stride) // Use for loop for clarity and potential optimization
    {
        in[globalIdx] += scalar; // Directly add scalar to improve readability
    }
}