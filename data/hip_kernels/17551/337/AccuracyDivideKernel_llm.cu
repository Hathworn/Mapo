#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use a single thread to perform division since it's a simple operation
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}