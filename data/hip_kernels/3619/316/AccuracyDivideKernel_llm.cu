#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use block-level atomic operation to safely update the accuracy
    atomicAdd(accuracy, -*accuracy / N);
}