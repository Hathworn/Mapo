#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cube(float * d_out, float * d_in) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    if(i < 96) {
        float val = d_in[i];
        d_out[i] = val * val * val;  // Compute cube using temporary variable
    }
}