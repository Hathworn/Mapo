#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
extern "C"
__global__ void slice_sparse_dense_nnz(double* inVal, int* inRowPtr, int* colInd, double* ret, int rl, int ru, int cl, int cu, int retClen) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if(tid >= inRowPtr[ru+1] - inRowPtr[rl]) return; // Early exit for out-of-range threads

    int i = tid + inRowPtr[rl];

    // Only slice if the index falls into the specified column range
    int colIndex = colInd[i];
    if(cl <= colIndex && colIndex <= cu) {
        // Efficient row index calculation
        int low = rl, high = ru, mid;
        while(low < high) {
            mid = (low + high) / 2;
            if(inRowPtr[mid+1] > i) 
                high = mid;
            else 
                low = mid + 1;
        }
        int rowIndex = low;
        ret[ (rowIndex - rl) * retClen + (colIndex - cl) ] = inVal[i];
    }
}