#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**********************************
When updating a kernel or adding a new one,
please compile the ptx file and commit it:
nvcc -ptx -arch=sm_30 SystemML.cu
***********************************/

/**
* Performs a slice operation where the input matrix is sparse and the output matrix is dense.
* This function avoids unnecessary sparse to dense conversion of the input matrix.
* Parallelization: rows of output matrix.
*
* @params inVal input val pointer
* @params inRowPtr input row pointer
* @params colInd input col index pointer
* @params ret dense output pointer
* @param rl row lower
* @param ru row upper
* @param cl column lower
* @param cu column upper
* @param retClen number of columns of output matrix
*/
extern "C"

/**
* Performs a slice operation where the input matrix is sparse and the output matrix is dense.
* This function avoids unnecessary sparse to dense conversion of the input matrix.
* Parallelization: subset of number of non-zeroes of input matrix.
*
* @params inVal input val pointer
* @params inRowPtr input row pointer
* @params colInd input col index pointer
* @params ret dense output pointer
* @param rl row lower
* @param ru row upper
* @param cl column lower
* @param cu column upper
* @param retClen number of columns of output matrix
*/
extern "C"

/**
* Performs a slice operation where the input matrix is dense and the output matrix is dense.
*
* @params in dense input pointer
* @params ret dense output pointer
* @param rl row lower
* @param ru row upper
* @param cl column lower
* @param cu column upper
* @param inClen number of columns of input matrix
* @param retRlen number of rows of output matrix
* @param retClen number of columns of output matrix
*/
extern "C"


/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/
extern "C"

extern "C"
__global__ void copy_u2l_dense(double* ret, int dim, int N) {
    // Efficiently map only the necessary threads
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    if (ix >= dim) return;
    
    for (int iy = ix + 1; iy < dim; ++iy) {
        int id_dest = iy * dim + ix;
        if (id_dest < N) {
            int id_src = ix * dim + iy;
            ret[id_dest] = ret[id_src];
        }
    }
}