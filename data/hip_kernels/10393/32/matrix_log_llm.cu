#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Licensed to the Apache Software Foundation (ASF) under one
* or more contributor license agreements.  See the NOTICE file
* distributed with this work for additional information
* regarding copyright ownership.  The ASF licenses this file
* to you under the Apache License, Version 2.0 (the
* "License"); you may not use this file except in compliance
* with the License.  You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing,
* software distributed under the License is distributed on an
* "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
* KIND, either express or implied.  See the License for the
* specific language governing permissions and limitations
* under the License.
*/

/**********************************
When updating a kernel or adding a new one,
please compile the ptx file and commit it:
nvcc -ptx -arch=sm_30 SystemML.cu
***********************************/

extern "C"
__global__ void matrix_log(double *A, double *C, unsigned int size) {
    // Optimize by using shared memory for faster access if block size is small
    __shared__ double sharedA[256]; 
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory before computation (assuming blockDim.x <= 256)
    if (index < size) {
        sharedA[threadIdx.x] = A[index];
        __syncthreads();

        // Perform log operation using shared memory
        C[index] = log(sharedA[threadIdx.x]);
    }
}