#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Licensed to the Apache Software Foundation (ASF) under one
* or more contributor license agreements.  See the NOTICE file
* distributed with this work for additional information
* regarding copyright ownership.  The ASF licenses this file
* to you under the Apache License, Version 2.0 (the
* "License"); you may not use this file except in compliance
* with the License.  You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing,
* software distributed under the License is distributed on an
* "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
* KIND, either express or implied.  See the License for the
* specific language governing permissions and limitations
* under the License.
*/

/**********************************
When updating a kernel or adding a new one,
please compile the ptx file and commit it:
nvcc -ptx -arch=sm_30 SystemML.cu
***********************************/

extern "C"
__global__ void slice_sparse_dense_row(double* inVal, int* inRowPtr, int* colInd, double* ret, int rl, int ru, int cl, int cu, int retClen) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int rowIndex = index + rl;
    
    // Ensure row index is within bounds
    if (rowIndex <= ru){
        // Cache read boundaries
        int start = inRowPtr[rowIndex];
        int end = inRowPtr[rowIndex + 1];
        
        // Loop over the non-zero elements in the sparse row
        for(int i = start; i < end; i++) {
            int colIdx = colInd[i];
            
            // Check if column index is within the slice range
            if(cl <= colIdx && colIdx <= cu) {
                ret[index * retClen + (colIdx - cl)] = inVal[i]; // Assign value to output
            }
        }
    }
}