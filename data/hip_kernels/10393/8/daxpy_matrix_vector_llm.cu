#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Licensed to the Apache Software Foundation (ASF) under one
* or more contributor license agreements.  See the NOTICE file
* distributed with this work for additional information
* regarding copyright ownership.  The ASF licenses this file
* to you under the Apache License, Version 2.0 (the
* "License"); you may not use this file except in compliance
* with the License.  You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing,
* software distributed under the License is distributed on an
* "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
* KIND, either express or implied.  See the License for the
* specific language governing permissions and limitations
* under the License.
*/

/**********************************
When updating a kernel or adding a new one,
please compile the ptx file and commit it:
nvcc -ptx -arch=sm_30 SystemML.cu
***********************************/

extern "C"
__global__ void daxpy_matrix_vector(double* A, double* B, double alpha, double* ret, int rlenA, int clenA, int rlenB, int clenB) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < rlenA * clenA) { // Check within total elements
        int ix = tid / clenA;
        int iy = tid % clenA;
        if (rlenB == 1) {
            ret[tid] = A[tid] + alpha * B[iy]; // Use tid directly for indexing
        } else {
            ret[tid] = A[tid] + alpha * B[ix]; // Use tid directly for indexing
        }
    }
}