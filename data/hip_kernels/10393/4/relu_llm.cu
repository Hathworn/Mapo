#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**
* Performs a slice operation where the input matrix is sparse and the output matrix is dense.
* This function avoids unnecessary sparse to dense conversion of the input matrix.
* Parallelization: rows of output matrix.
*
* @params inVal input val pointer
* @params inRowPtr input row pointer
* @params colInd input col index pointer
* @params ret dense output pointer
* @param rl row lower
* @param ru row upper
* @param cl column lower
* @param cu column upper
* @param retClen number of columns of output matrix
*/
extern "C"

/**
* Performs a slice operation where the input matrix is sparse and the output matrix is dense.
* This function avoids unnecessary sparse to dense conversion of the input matrix.
* Parallelization: subset of number of non-zeroes of input matrix.
*
* @params inVal input val pointer
* @params inRowPtr input row pointer
* @params colInd input col index pointer
* @params ret dense output pointer
* @param rl row lower
* @param ru row upper
* @param cl column lower
* @param cu column upper
* @param retClen number of columns of output matrix
*/
extern "C"

/**
* Performs a slice operation where the input matrix is dense and the output matrix is dense.
*
* @params in dense input pointer
* @params ret dense output pointer
* @param rl row lower
* @param ru row upper
* @param cl column lower
* @param cu column upper
* @param inClen number of columns of input matrix
* @param retRlen number of rows of output matrix
* @param retClen number of columns of output matrix
*/
extern "C"


/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/
extern "C"

extern "C"
__global__ void relu(double* A, double* ret, int rlen, int clen) {
    // Compute unique thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Compute row and column indices
    int ix = tid / clen;
    int iy = tid % clen;

    // Perform ReLU operation only within matrix bounds
    if (ix < rlen && iy < clen) {
        ret[tid] = max(0.0, A[tid]);
    }
}