#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/

extern "C"
__global__ void relu_backward(double* X, double* dout, double* ret, int rlen, int clen) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;
    
    // Process multiple elements per thread by striding through the data
    for (int i = tid; i < rlen * clen; i += totalThreads) {
        ret[i] = X[i] > 0 ? dout[i] : 0; // Compute ReLU backward
    }
}