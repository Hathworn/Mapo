#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function for matrix ceil operation.
extern "C"
__global__ void matrix_ceil(double *A, double *C, unsigned int size) {
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop unrolling: Process multiple elements per loop iteration
    for (int i = index; i < size; i += blockDim.x * gridDim.x) {
        C[i] = ceil(A[i]);
    }
}