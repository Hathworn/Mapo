#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void matrix_sinh(double *A, double *C, unsigned int size) {
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds and perform operation
    if (index < size) {
        // Use inline function to improve performance
        C[index] = __sinh(A[index]);
    }
}