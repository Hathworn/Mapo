#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void polynomial_expansion (float* poly, int degree, int n, float* array)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
    {
        float val = poly[0]; // Initialize with the first coefficient
        float exp = array[idx]; // Compute initial x
        for (int x = 1; x <= degree; ++x)
        {
            val += exp * poly[x]; // Multiply and accumulate
            exp *= array[idx]; // Increment power of x
        }
        array[idx] = val;
    }
}