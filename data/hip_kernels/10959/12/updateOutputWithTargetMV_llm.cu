#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateOutputWithTargetMV(const float* input, const float* weight, const float* bias, const float* mapping, const float* n_class_in_cluster, const float* class_start_indices, const float* target, const long input_stride0, const long weight_stride0, const long score_stride0, long input_size, float* score) {
    __shared__ float buffer[MV_BUFFER_SIZE];
    
    // Align input and score to current sample in minibatch
    input += input_stride0 * blockIdx.y;
    score += score_stride0 * blockIdx.y;

    // Get the indices corresponding to the target
    const int itarget = (int)(target[blockIdx.y] - 0.5f);
    const int cluster_target = (int)(mapping[2 * itarget] - 0.5f);
    const int iclass_start = (int)(class_start_indices[cluster_target] + 0.5f);
    const int cluster_size = (int)(n_class_in_cluster[cluster_target] + 0.5f);

    // Initialize thread and block indices
    const int lineIdx = blockIdx.x;
    const int nLinesParallel = gridDim.x;
    const int tidxx = threadIdx.x;

    // Matrix vector multiplication
    for (int iline = lineIdx; iline < cluster_size; iline += nLinesParallel) {
        const float* weight0 = weight + weight_stride0 * (iclass_start + iline);
        
        __syncthreads();
        register float tmp = 0.f;
        
        // Calculate dot product of input and weight
        for (int i = tidxx; i < input_size; i += MV_BUFFER_SIZE) {
            tmp += input[i] * weight0[i];
        }
        buffer[tidxx] = tmp;

        // Reduce buffer to compute final score
        __syncthreads();
        tmp = 0.f;
        if (tidxx < MV_BUFFER_SIZE / MV_N_REDUCE) {
            for (int i = tidxx * MV_N_REDUCE; i < (tidxx + 1) * MV_N_REDUCE; ++i) {
                tmp += buffer[i];
            }
            buffer[tidxx] = tmp;
        }
        __syncthreads();

        // Store result
        if (tidxx == 0) {
            tmp = buffer[0];
            #pragma unroll
            for (int i = 1; i < MV_BUFFER_SIZE / MV_N_REDUCE; ++i) {
                tmp += buffer[i];
            }
            score[iline] = tmp + bias[iclass_start + iline];
        }
    }
}