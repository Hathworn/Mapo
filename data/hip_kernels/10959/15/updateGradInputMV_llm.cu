#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateGradInputMV(const float* score, const float* weight, const float* mapping, const float* n_class_in_cluster, const float* class_start_indices, const float* target, const long gradInput_stride0, const long weight_stride0, const long score_stride0, int input_size, float* gradInput) {
    // align input and score to current sample in minibatch
    gradInput += gradInput_stride0 * blockIdx.y;
    score += score_stride0 * blockIdx.y;

    // get the indices corresponding to the target
    const int itarget = (int)(target[blockIdx.y] - 0.5f); // -0.5: 1-based to 0-based
    const int cluster_target = (int)(mapping[2 * itarget] - 0.5f);
    const int iclass_start = (int)(class_start_indices[cluster_target] + 0.5f);
    const int cluster_size = (int)(n_class_in_cluster[cluster_target] + 0.5f);

    // get the start index for weight of the target cluster
    weight += weight_stride0 * iclass_start;

    // calculate column index of the thread
    const int colIdx = blockIdx.x * blockDim.x + threadIdx.x;
    const int nColParallel = gridDim.x * blockDim.x;

    // loop over columns this thread needs to process
    for (int icol = colIdx; icol < input_size; icol += nColParallel) {
        // get the starting address of the weights for the current column
        const float* weight0 = weight + icol;
        
        // accumulate score * weight
        float tmp = 0.f;
        for (int i = 0; i < cluster_size; ++i) {
            tmp += score[i] * weight0[weight_stride0 * i];
        }
        
        // write the result to gradInput
        gradInput[icol] = tmp;
    }
}