#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TemporalConvolutionTBC_bp_bias(float* matrix, float* target, int rows, int stride, float scale) {
    // Use shared memory for partial sums
    __shared__ float sharedData[32];
    
    int i = blockIdx.x * blockDim.x + threadIdx.x; // More flexible for varying block sizes
    float t = 0;

    // Each block processes multiple rows and accumulates results
    for (int j = blockIdx.y; j < rows; j += gridDim.y) {
        t += matrix[j * stride + i];
    }

    // Store the partial sum to shared memory
    sharedData[threadIdx.x] = t;

    // Synchronize threads within the block
    __syncthreads();

    // Sum the partial results in shared memory
    if (threadIdx.x == 0) {
        float blockSum = 0;
        for (int k = 0; k < blockDim.x; ++k) {
            blockSum += sharedData[k];
        }
        // Use atomic add to accumulate the final results from each block
        atomicAdd(&target[blockIdx.x * blockDim.x], blockSum * scale);
    }
}