#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateGradInputLSM(const float* target, const float* mapping, const float* n_class_in_cluster, float* class_score, float* class_logsum, float* cluster_score, float* cluster_logsum, const long class_score_stride0, const long cluster_score_stride0, int n_clusters) {
    const int tidx = threadIdx.x;
    const int nthreads = blockDim.x;

    const int itarget = static_cast<int>(target[blockIdx.x] - 0.5f);      // Enhanced precision.
    const int cluster_target = static_cast<int>(mapping[2*itarget] - 0.5f);
    const int idx_in_cluster_target = static_cast<int>(mapping[2*itarget+1] - 0.5f);
    const int cluster_size = static_cast<int>(n_class_in_cluster[cluster_target] + 0.5f);

    float* score;
    float logsum_k;
    float* target_score;
    int N;

    if (blockIdx.y == 0) {
        score = cluster_score + blockIdx.x * cluster_score_stride0;
        logsum_k = cluster_logsum[blockIdx.x];
        N = n_clusters;
        target_score = score + cluster_target;
    } else {
        score = class_score + blockIdx.x * class_score_stride0;
        logsum_k = class_logsum[blockIdx.x];
        N = cluster_size;
        target_score = score + idx_in_cluster_target;
    }

    // Parallelized score updates
    for (int i = tidx; i < N; i += nthreads) {
        score[i] = expf(score[i] - logsum_k);    // Apply exponentiation and normalization.
    }
    __syncthreads();

    // Single thread adjustment for the target score
    if (tidx == 0) {
        atomicAdd(target_score, -1.0f);          // Atomic operation to avoid race conditions.
    }
}