#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TestDimensions()
{
    int const num_threads_per_block = blockDim.x * blockDim.y * blockDim.z;
    int const num_blocks = gridDim.x * gridDim.y * gridDim.z;

    // Calculate thread_id using efficient arithmetic and memory operations
    int const thread_id = threadIdx.x + 
                          threadIdx.y * blockDim.x + 
                          threadIdx.z * blockDim.x * blockDim.y + 
                          (blockIdx.x + 
                          blockIdx.y * gridDim.x + 
                          blockIdx.z * gridDim.x * gridDim.y) * num_threads_per_block;

    if(thread_id == 0)
    {
        printf("gridDim   = x: %6d / y: %6d / z: %6d\r\n", gridDim.x, gridDim.y, gridDim.z);
        printf("blockDim  = x: %6d / y: %6d / z: %6d\r\n", blockDim.x, blockDim.y, blockDim.z);
        printf("num_threads_per_block: %6d\r\n", num_threads_per_block);
        printf("num_blocks           : %6d\r\n", num_blocks);
        printf("threads_stride       : %6d\r\n", num_threads_per_block * num_blocks);
    }

    printf("tidx | %6d | %6d | %6d | bidx | %6d | %6d | %6d | "
           "gdim | %6d | %6d | %6d | bdim | %6d | %6d | %6d | "
           "thread_id | %6d |\r\n",
           threadIdx.x, threadIdx.y, threadIdx.z,
           blockIdx.x, blockIdx.y, blockIdx.z,
           gridDim.x, gridDim.y, gridDim.z,
           blockDim.x, blockDim.y, blockDim.z, thread_id);
}