#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "change_theta_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    const int ncoord = 1;
const float3 *theta = NULL;
hipMalloc(&theta, XSIZE*YSIZE);
float4 *thetax = NULL;
hipMalloc(&thetax, XSIZE*YSIZE);
float4 *thetay = NULL;
hipMalloc(&thetay, XSIZE*YSIZE);
float4 *thetaz = NULL;
hipMalloc(&thetaz, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
change_theta<<<gridBlock, threadBlock>>>(ncoord,theta,thetax,thetay,thetaz);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
change_theta<<<gridBlock, threadBlock>>>(ncoord,theta,thetax,thetay,thetaz);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
change_theta<<<gridBlock, threadBlock>>>(ncoord,theta,thetax,thetay,thetaz);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}