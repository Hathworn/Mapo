#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void change_theta(const int ncoord, const float3 *theta, float4 *thetax, float4 *thetay, float4 *thetaz) {
    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Use strided loop for coalesced access and to handle large ncoord
    for (unsigned int i = pos; i < ncoord; i += gridDim.x * blockDim.x) {
        const float3 t0 = theta[i * 4];
        const float3 t1 = theta[i * 4 + 1];
        const float3 t2 = theta[i * 4 + 2];
        const float3 t3 = theta[i * 4 + 3];

        thetax[i] = make_float4(t0.x, t1.x, t2.x, t3.x); // Pack the x components
        thetay[i] = make_float4(t0.y, t1.y, t2.y, t3.y); // Pack the y components
        thetaz[i] = make_float4(t0.z, t1.z, t2.z, t3.z); // Pack the z components
    }
}