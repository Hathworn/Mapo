#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "fill_bspline_4.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    const float4 *xyzq = NULL;
hipMalloc(&xyzq, XSIZE*YSIZE);
const int ncoord = 1;
const float *recip = NULL;
hipMalloc(&recip, XSIZE*YSIZE);
const int nfftx = 1;
const int nffty = 1;
const int nfftz = 1;
int *gix = NULL;
hipMalloc(&gix, XSIZE*YSIZE);
int *giy = NULL;
hipMalloc(&giy, XSIZE*YSIZE);
int *giz = NULL;
hipMalloc(&giz, XSIZE*YSIZE);
float *charge = NULL;
hipMalloc(&charge, XSIZE*YSIZE);
float *thetax = NULL;
hipMalloc(&thetax, XSIZE*YSIZE);
float *thetay = NULL;
hipMalloc(&thetay, XSIZE*YSIZE);
float *thetaz = NULL;
hipMalloc(&thetaz, XSIZE*YSIZE);
float *dthetax = NULL;
hipMalloc(&dthetax, XSIZE*YSIZE);
float *dthetay = NULL;
hipMalloc(&dthetay, XSIZE*YSIZE);
float *dthetaz = NULL;
hipMalloc(&dthetaz, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
fill_bspline_4<<<gridBlock, threadBlock>>>(xyzq,ncoord,recip,nfftx,nffty,nfftz,gix,giy,giz,charge,thetax,thetay,thetaz,dthetax,dthetay,dthetaz);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
fill_bspline_4<<<gridBlock, threadBlock>>>(xyzq,ncoord,recip,nfftx,nffty,nfftz,gix,giy,giz,charge,thetax,thetay,thetaz,dthetax,dthetay,dthetaz);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
fill_bspline_4<<<gridBlock, threadBlock>>>(xyzq,ncoord,recip,nfftx,nffty,nfftz,gix,giy,giz,charge,thetax,thetay,thetaz,dthetax,dthetay,dthetaz);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}