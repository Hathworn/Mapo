#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_bspline_4(const float4 *xyzq, const int ncoord, const float *recip, const int nfftx, const int nffty, const int nfftz, int *gix, int *giy, int *giz, float *charge, float *thetax, float *thetay, float *thetaz, float *dthetax, float *dthetay, float *dthetaz) {

    unsigned int pos = blockIdx.x * blockDim.x + threadIdx.x;

    while (pos < ncoord) {
        float4 xyzqi = xyzq[pos];
        float x = xyzqi.x;
        float y = xyzqi.y;
        float z = xyzqi.z;
        float q = xyzqi.w;

        // Precomputed
        float recip_x = recip[0];
        float recip_y = recip[1];
        float recip_z = recip[2];
        float recip_3x = recip[3];
        float recip_3y = recip[4];
        float recip_3z = recip[5];
        float recip_6x = recip[6];
        float recip_6y = recip[7];
        float recip_6z = recip[8];

        float w = x * recip_x + y * recip_y + z * recip_z + 2.0f;
        float frx = nfftx * (w - (floorf(w + 0.5f) - 0.5f));

        w = x * recip_3x + y * recip_3y + z * recip_3z + 2.0f;
        float fry = nffty * (w - (floorf(w + 0.5f) - 0.5f));

        w = x * recip_6x + y * recip_6y + z * recip_6z + 2.0f;
        float frz = nfftz * (w - (floorf(w + 0.5f) - 0.5f));

        int frxi = static_cast<int>(frx);
        int fryi = static_cast<int>(fry);
        int frzi = static_cast<int>(frz);

        float wx = frx - frxi;
        float wy = fry - fryi;
        float wz = frz - frzi;

        gix[pos] = frxi;
        giy[pos] = fryi;
        giz[pos] = frzi;
        charge[pos] = q;

        float one_minus_wx = 1.0f - wx;
        float one_minus_wy = 1.0f - wy;
        float one_minus_wz = 1.0f - wz;

        float3 theta_tmp[4];
        float3 dtheta_tmp[4];

        theta_tmp[3] = make_float3(0.0f, 0.0f, 0.0f);
        theta_tmp[1] = make_float3(wx, wy, wz);
        theta_tmp[0] = make_float3(one_minus_wx, one_minus_wy, one_minus_wz);

        // B-spline recursion
        theta_tmp[2] = make_float3(0.5f * wx * theta_tmp[1].x,
                                   0.5f * wy * theta_tmp[1].y,
                                   0.5f * wz * theta_tmp[1].z);

        theta_tmp[1] = make_float3(0.5f * ((1.0f + wx) * theta_tmp[0].x + (2.0f - wx) * theta_tmp[1].x),
                                   0.5f * ((1.0f + wy) * theta_tmp[0].y + (2.0f - wy) * theta_tmp[1].y),
                                   0.5f * ((1.0f + wz) * theta_tmp[0].z + (2.0f - wz) * theta_tmp[1].z));

        theta_tmp[0] = make_float3(0.5f * one_minus_wx * theta_tmp[0].x,
                                   0.5f * one_minus_wy * theta_tmp[0].y,
                                   0.5f * one_minus_wz * theta_tmp[0].z);

        // Differentiation
        dtheta_tmp[0] = make_float3(-theta_tmp[0].x, -theta_tmp[0].y, -theta_tmp[0].z);
        dtheta_tmp[1] = make_float3(theta_tmp[0].x - theta_tmp[1].x, theta_tmp[0].y - theta_tmp[1].y, theta_tmp[0].z - theta_tmp[1].z);
        dtheta_tmp[2] = make_float3(theta_tmp[1].x - theta_tmp[2].x, theta_tmp[1].y - theta_tmp[2].y, theta_tmp[1].z - theta_tmp[2].z);
        dtheta_tmp[3] = make_float3(theta_tmp[2].x - theta_tmp[3].x, theta_tmp[2].y - theta_tmp[3].y, theta_tmp[2].z - theta_tmp[3].z);

        // Final recursion
        theta_tmp[3] = make_float3((1.0f / 3.0f) * wx * theta_tmp[2].x,
                                   (1.0f / 3.0f) * wy * theta_tmp[2].y,
                                   (1.0f / 3.0f) * wz * theta_tmp[2].z);

        theta_tmp[2] = make_float3((1.0f / 3.0f) * ((1.0f + wx) * theta_tmp[1].x + (3.0f - wx) * theta_tmp[2].x),
                                   (1.0f / 3.0f) * ((1.0f + wy) * theta_tmp[1].y + (3.0f - wy) * theta_tmp[2].y),
                                   (1.0f / 3.0f) * ((1.0f + wz) * theta_tmp[1].z + (3.0f - wz) * theta_tmp[2].z));

        theta_tmp[1] = make_float3((1.0f / 3.0f) * ((2.0f + wx) * theta_tmp[0].x + (2.0f - wx) * theta_tmp[1].x),
                                   (1.0f / 3.0f) * ((2.0f + wy) * theta_tmp[0].y + (2.0f - wy) * theta_tmp[1].y),
                                   (1.0f / 3.0f) * ((2.0f + wz) * theta_tmp[0].z + (2.0f - wz) * theta_tmp[1].z));

        theta_tmp[0] = make_float3((1.0f / 3.0f) * one_minus_wx * theta_tmp[0].x,
                                   (1.0f / 3.0f) * one_minus_wy * theta_tmp[0].y,
                                   (1.0f / 3.0f) * one_minus_wz * theta_tmp[0].z);

        int pos4 = pos * 4;

        // Store into global memory
        thetax[pos4] = theta_tmp[0].x;
        thetax[pos4 + 1] = theta_tmp[1].x;
        thetax[pos4 + 2] = theta_tmp[2].x;
        thetax[pos4 + 3] = theta_tmp[3].x;

        thetay[pos4] = theta_tmp[0].y;
        thetay[pos4 + 1] = theta_tmp[1].y;
        thetay[pos4 + 2] = theta_tmp[2].y;
        thetay[pos4 + 3] = theta_tmp[3].y;

        thetaz[pos4] = theta_tmp[0].z;
        thetaz[pos4 + 1] = theta_tmp[1].z;
        thetaz[pos4 + 2] = theta_tmp[2].z;
        thetaz[pos4 + 3] = theta_tmp[3].z;

        dthetax[pos4] = dtheta_tmp[0].x;
        dthetax[pos4 + 1] = dtheta_tmp[1].x;
        dthetax[pos4 + 2] = dtheta_tmp[2].x;
        dthetax[pos4 + 3] = dtheta_tmp[3].x;

        dthetay[pos4] = dtheta_tmp[0].y;
        dthetay[pos4 + 1] = dtheta_tmp[1].y;
        dthetay[pos4 + 2] = dtheta_tmp[2].y;
        dthetay[pos4 + 3] = dtheta_tmp[3].y;

        dthetaz[pos4] = dtheta_tmp[0].z;
        dthetaz[pos4 + 1] = dtheta_tmp[1].z;
        dthetaz[pos4 + 2] = dtheta_tmp[2].z;
        dthetaz[pos4 + 3] = dtheta_tmp[3].z;

        pos += blockDim.x * gridDim.x;
    }
}