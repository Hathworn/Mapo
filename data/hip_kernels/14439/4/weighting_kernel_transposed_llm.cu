#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

#define IDX2C(i, j, ld) ((j)*(ld)+(i))
#define SQR(x) ((x)*(x)) // x^2

__global__ void weighting_kernel_transposed(double const* matrices, double const* weights, double* results) {
    // Optimize index calculation using built-in variables
    int matrix_index = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    int weight_index = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform weighted multiplication
    results[matrix_index] = matrices[matrix_index] * weights[weight_index];
}