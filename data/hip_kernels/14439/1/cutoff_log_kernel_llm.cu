#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define IDX2C(i, j, ld) ((j)*(ld)+(i))
#define SQR(x)      ((x)*(x))                        // x^2

__global__ void cutoff_log_kernel(double* device_array, double min_signal) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    double value = device_array[thread_id];
    // Use ternary operator for concise conditional assignment
    device_array[thread_id] = log(value < min_signal ? min_signal : value);
}