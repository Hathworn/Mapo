#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define IDX2C(i, j, ld) ((j)*(ld)+(i))
#define SQR(x)      ((x)*(x))                        // x^2

__global__ void transpose_kernel(double const* matrices, double* transposed) {
    // Improve memory access by reordering the thread indices 
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = threadIdx.y;
    int matrix_index = IDX2C(row, col, blockDim.y);
    int transpose_index = IDX2C(col, row, blockDim.x);
    transposed[transpose_index] = matrices[matrix_index];
}