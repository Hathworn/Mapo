#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define IDX2C(i, j, ld) ((j)*(ld)+(i))
#define SQR(x)      ((x)*(x))                        // x^2

__global__ void assemble_tensors(double const* tensor_input, double* tensors, int tensor_input_elements) {
    int tensor_matrix_offset = blockIdx.x * TENSOR_DIMENSIONS * TENSOR_DIMENSIONS;
    int input_matrix_offset = blockIdx.x * tensor_input_elements;

    // Use shared memory for input values to reduce global memory access
    __shared__ double input_vals[6];
    input_vals[0] = tensor_input[input_matrix_offset + 0];
    input_vals[1] = tensor_input[input_matrix_offset + 1];
    input_vals[2] = tensor_input[input_matrix_offset + 2];
    input_vals[3] = tensor_input[input_matrix_offset + 3];
    input_vals[4] = tensor_input[input_matrix_offset + 4];
    input_vals[5] = tensor_input[input_matrix_offset + 5];
    __syncthreads();

    // Writing results using shared memory values
    tensors[tensor_matrix_offset + 0] = input_vals[0];
    tensors[tensor_matrix_offset + 1] = input_vals[1];
    tensors[tensor_matrix_offset + 2] = input_vals[3];
    tensors[tensor_matrix_offset + 3] = input_vals[1];
    tensors[tensor_matrix_offset + 4] = input_vals[2];
    tensors[tensor_matrix_offset + 5] = input_vals[4];
    tensors[tensor_matrix_offset + 6] = input_vals[3];
    tensors[tensor_matrix_offset + 7] = input_vals[4];
    tensors[tensor_matrix_offset + 8] = input_vals[5];
}