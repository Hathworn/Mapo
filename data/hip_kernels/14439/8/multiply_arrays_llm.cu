#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define IDX2C(i, j, ld) ((j)*(ld)+(i))
#define SQR(x)      ((x)*(x))                        // x^2

__global__ void multiply_arrays(double* signals, double const* weights) {
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform the multiplication if within bounds
    if (idx < N) {
        signals[idx] *= weights[idx];
    }
}
```
