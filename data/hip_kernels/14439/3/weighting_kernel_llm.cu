#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define IDX2C(i, j, ld) ((j)*(ld)+(i))
#define SQR(x)      ((x)*(x))                        // x^2

__global__ void weighting_kernel(double const* matrices, double const* weights, double* results) {
    // Using shared memory to cache weights for improved performance
    __shared__ double shared_weights[blockDim.y];
    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;
    int block_index = blockDim.y * tid_x + tid_y;
    int matrix_index = blockIdx.x * blockDim.x * blockDim.y + block_index;
    
    // Load weights into shared memory
    if (tid_y == 0) {
        shared_weights[tid_x] = weights[blockIdx.x * blockDim.y + tid_x];
    }
    __syncthreads();

    // Compute result with cached weights
    results[matrix_index] = matrices[block_index] * shared_weights[tid_y];
}