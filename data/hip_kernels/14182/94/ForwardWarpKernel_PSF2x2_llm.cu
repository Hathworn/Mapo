#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Precompute values for offsets with time scaling
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // Compute pixel boundary and deltas
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    // Convert to integer
    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];

    // Declare weight outside to avoid redeclaration
    float weight;

    // Combined condition check with addition to improve performance
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0))
    {
        weight = dx * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Update tx and recompute if within bounds
    tx -= 1;
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0))
    {
        weight = (1.0f - dx) * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Update ty and recompute if within bounds
    ty -= 1;
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0))
    {
        weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Update tx and recompute if within bounds
    tx += 1;
    if (!(tx >= w || tx < 0 || ty >= h || ty < 0))
    {
        weight = dx * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}
``` 