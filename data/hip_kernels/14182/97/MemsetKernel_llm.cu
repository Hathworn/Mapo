#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Pre-compute global indices
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds
    if (i < h && j < w) {
        // Compute linear index and set value
        image[i * w + j] = value;
    }
}