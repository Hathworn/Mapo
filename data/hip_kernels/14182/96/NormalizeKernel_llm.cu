#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Early exit for out-of-bounds threads
    if (i >= h || j >= w) return;

    const int pos = i * s + j;

    // Load scale and compute inverse only once; avoid branching
    float scale = normalization_factor[pos];
    float invScale = (scale == 0.0f) ? 1.0f : __fdividef(1.0f, scale);

    image[pos] *= invScale;
}