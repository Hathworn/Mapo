#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate the point index for the current thread
    int ptidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop for better utilization of GPU resources
    while (ptidx < npoints) 
    {
        short2 loc = loc_[ptidx];

        // Apply scale transformation
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
        
        // Increment by total number of threads in grid
        ptidx += blockDim.x * gridDim.x;
    }
}