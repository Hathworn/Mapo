#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Translate index inversion with offsets
__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

// Translate index
__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

// Calculate index with nc, height, width
__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x;
}

// Optimized global kernel function
__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * blockDim.x * gridDim.x;
    if (ii >= no_elements) return;

    // Efficiently loop through scale factors using pre-calculated indices
    const int targetIdxBase = translate_idx(ii, d1, d2, d3, scale_factor);
    for (int i = 0; i < scale_factor; i++) {
        for (int j = 0; j < scale_factor; j++) {
            int ipidx = targetIdxBase + translate_idx_inv(0, 1, 1, d3, scale_factor, i, j);
            gradInput_data[ii] += gradOutput_data[ipidx];
        }
    }
}