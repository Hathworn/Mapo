#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Calculate new index by considering multiple threads
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Ensure the operation is performed by only one thread
    if (idx == 0) {
        *accuracy /= N;
    }
}