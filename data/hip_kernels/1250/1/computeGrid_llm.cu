#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZE 30000				//Length and width of inner grid in threads
#define DIM (SIZE + 2)			//Length and width of the entire grid in threads
#define GRID_SIZE 1500 			//Length and width of inner grid in blocks
#define BLOCK_SIZE 20 			//Length and width of block in threads
#define MEM_SIZE (sizeof(float) * DIM * DIM)
#define TIME_STEPS 1
#define PINNED 0

void fillGrid(float* grid);

__global__ void computeGrid(float* read, float* write) {
    // Use shared memory for faster access
    __shared__ float shBlock[BLOCK_SIZE + 2][BLOCK_SIZE + 2];

    // Calculate global and local indices
    int x = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int y = blockDim.y * blockIdx.y + threadIdx.y + 1;
    int localX = threadIdx.x + 1;
    int localY = threadIdx.y + 1;

    // Load the main cell and four neighbors into shared memory
    shBlock[localY][localX] = read[DIM * y + x];
    if (threadIdx.x == 0) shBlock[localY][0] = read[DIM * y + x - 1];
    if (threadIdx.x == blockDim.x - 1) shBlock[localY][localX + 1] = read[DIM * y + x + 1];
    if (threadIdx.y == 0) shBlock[0][localX] = read[DIM * (y - 1) + x];
    if (threadIdx.y == blockDim.y - 1) shBlock[localY + 1][localX] = read[DIM * (y + 1) + x];
    
    // Synchronize to ensure all threads have loaded data
    __syncthreads();

    // Compute the result using shared memory
    if (threadIdx.x != 0 && threadIdx.x != blockDim.x - 1 && threadIdx.y != 0 && threadIdx.y != blockDim.y - 1) {
        write[DIM * y + x] = shBlock[localY - 1][localX] + shBlock[localY + 1][localX] +
                             shBlock[localY][localX - 1] + shBlock[localY][localX + 1];
    }
}