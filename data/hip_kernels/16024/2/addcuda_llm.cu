#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

#define D 3
#define N 200
#define K 512
#define Nt 20
#define Rt 0.1f
#define c 0.001f
#define ct 0.0001f

__global__ void addcuda(float* Q, float* P, float* Qt, float* Pt, float* Eg, float* Epg) {
    int x = blockIdx.x;
    int y = threadIdx.x;
    int i = x * K * D + y * D;

    // Load position values
    float Qx = Q[i];
    float Qy = Q[i + 1];
    float Qz = Q[i + 2];

    // Load momentum values
    float Px = P[i];
    float Py = P[i + 1];
    float Pz = P[i + 2];
    
    // Load energy values
    float E = Eg[i / 3];
    float Ep = Epg[i / 3];

    // Calculate new position
    float nQx = Qx + c * Px;
    float nQy = Qy + c * Py;
    float nQz = Qz + c * Pz;

    // Reflect from boundaries
    if ((nQx > 1.0f) || (nQx < 0.0f)) Px = -Px;
    if ((nQy > 1.0f) || (nQy < 0.0f)) Py = -Py;
    if ((nQz > 1.0f) || (nQz < 0.0f)) Pz = -Pz;

    // Reflect from turbulences
    for (int nt = 0; nt < Nt; nt++) {
        float QtX = Qt[nt];
        float QtY = Qt[nt + 1];
        float QtZ = Qt[nt + 2];

        float Range = sqrtf(powf(Qx - QtX, 2) + powf(Qy - QtY, 2) + powf(Qz - QtZ, 2));
        float nRange = sqrtf(powf(nQx - QtX, 2) + powf(nQy - QtY, 2) + powf(nQz - QtZ, 2));

        if ((Range > Rt) && (nRange < Rt)) {
            float DirX = (nQx - QtX) / Range;
            float DirY = (nQy - QtY) / Range;
            float DirZ = (nQz - QtZ) / Range;
            float PnormKoe = Px * DirX + Py * DirY + Pz * DirZ;
            float Pnormt = Pt[nt] * DirX + Pt[nt + 1] * DirY + Pt[nt + 2] * DirZ;
            E -= (ct / c) * PnormKoe * PnormKoe * Pnormt * fabs(Pnormt);
            Px -= 2.0f * DirX;
            Py -= 2.0f * DirY;
            Pz -= 2.0f * DirZ;
        }
    }

    // Particle behavior and energy adjustment
    if ((nQz > 1.0f) && (E > Ep)) E = 100.0f;
    if (nQz > 0.5f) E -= 0.0001f;
    if (((nQz > 0.5f) && (Qz < 0.5f)) || ((Qz > 0.5f) && (nQz < 0.5f))) E += 1.0f;

    // Write back to memory
    Q[i] = nQx;
    Q[i + 1] = nQy;
    Q[i + 2] = nQz;

    P[i] = Px;
    P[i + 1] = Py;
    P[i + 2] = Pz;
    Eg[i / 3] = E;
}