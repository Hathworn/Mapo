#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

#define D 3
#define N 200
#define K 512
#define Nt 20
#define Rt 0.1f
#define c 0.001f
#define ct 0.0001f

__global__ void NextQTur(float* Qt, float* Pt) {
    int i = threadIdx.x;
    int index = i * D; // Calculate the starting index for the current thread
    // Update Qt with Pt values scaled by ct
    Qt[index] += Pt[index] * ct;
    Qt[index + 1] += Pt[index + 1] * ct;
    Qt[index + 2] += Pt[index + 2] * ct;
}