#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

#define D 3
#define N 200
#define K 512
#define Nt 20
#define Rt 0.1f
#define c 0.001f
#define ct 0.0001f

__global__ void Sqrt(float* Q, float* P, float* Qt, float* Pt, float* Eg, float* Epg) {
    int x = blockIdx.x;
    int y = threadIdx.x;
    int i = x * K * D + y * D;
    
    // Optimization: Unroll the loop manually for efficiency
    Q[i] = 0.01f;
    Qt[i] = 0.6f;
    P[i] = 0.3f;
    Pt[i] = 0.5f;
    
    Q[i + 1] = 0.01f;
    Qt[i + 1] = 0.6f;
    P[i + 1] = 0.3f;
    Pt[i + 1] = 0.5f;
    
    Q[i + 2] = 0.01f;
    Qt[i + 2] = 0.6f;
    P[i + 2] = 0.3f;
    Pt[i + 2] = 0.5f;
    
    // Reduce redundant calculations by directly accessing the index
    Epg[i / D] = 100000;
    Eg[i / D] = 0.5f;
}