#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to divide each accuracy element by N using a single thread
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Perform divide operation using thread 0 only
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}