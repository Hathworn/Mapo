#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    const uint numEls = tgtWidth * tgtHeight;

    // Loop over elements this thread should process, optimizing by unrolling
    for (uint i = idx; i < numEls; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}