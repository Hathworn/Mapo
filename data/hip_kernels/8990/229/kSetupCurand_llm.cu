#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate thread index
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    // Initialize hiprand state with unique seed per thread
    hiprand_init(seed, tidx, 0, &state[tidx]);
}