#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define G_CONSTANT 6.67430e-11  // Define gravitational constant

__global__ void forces_and_step(double *bodies_input, double *bodies_output, unsigned int count, double dt, uint16_t bods_per_thread)
{
    unsigned int index = ((blockIdx.x * blockDim.x) + threadIdx.x) * 5 * bods_per_thread;

    for (int b = 0; b < bods_per_thread; b++)
    {
        // If there are more threads than data discard the extra computations to stay in defined memory
        if (index < count * 5)
        {
            // Calculate force for this particle
            double fx = 0.0;
            double fy = 0.0;
            double m1 = bodies_input[index + 4];  // Mass of current body

            for (uint32_t i = 0; i < count * 5; i += 5)
            {
                double dir_x = bodies_input[i] - bodies_input[index];
                double dir_y = bodies_input[i + 1] - bodies_input[index + 1];
                double m2 = bodies_input[i + 4];  // Mass of other body

                // Make sure there is no division by zero
                double distance_squared = dir_x * dir_x + dir_y * dir_y;
                if (distance_squared > 0.0)
                {
                    double distance = sqrt(distance_squared);
                    double force_magnitude = G_CONSTANT * m1 * m2 / (distance_squared * distance);
                    fx += dir_x * force_magnitude;
                    fy += dir_y * force_magnitude;
                }
            }

            // Integration
            bodies_output[index + 2] = bodies_input[index + 2] + (fx / m1) * dt;
            bodies_output[index + 3] = bodies_input[index + 3] + (fy / m1) * dt;
            bodies_output[index] = bodies_input[index] + bodies_output[index + 2] * dt;
            bodies_output[index + 1] = bodies_input[index + 1] + bodies_output[index + 3] * dt;
            bodies_output[index + 4] = m1;  // Preserve the mass
        }
        index += 5;
    }
}