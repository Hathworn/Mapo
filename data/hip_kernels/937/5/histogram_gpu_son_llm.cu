#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogram_gpu_son(unsigned char * d_img, unsigned int * d_hist, int img_size, int serialNum)
{
    // Use shared memory to reduce global memory access latency
    extern __shared__ unsigned int aa[];
    int x = threadIdx.x + blockDim.x * blockIdx.x;

    // Initialize shared memory
    for (int i = 0; i < ROLLSIZE; i++) {
        aa[(i << 8) + threadIdx.x] = 0;
    }
    __syncthreads();

    // Determine the range for processing based on thread index and serialNum
    int end = (x + 1) * serialNum;
    if (end >= img_size) end = img_size;

    // Fill shared memory with histogram data
    for (int i = x * serialNum; i < end; i++) {
        atomicAdd(&(aa[((threadIdx.x >> 4) << 8) + d_img[i]]), 1);
    }
    __syncthreads();

    // Reduce shared memory data within the block
    unsigned int s;
    for (s = 16 / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            for (int i = 0; i < 256; i++) {
                aa[(i << 8) + threadIdx.x] += aa[((i + s) << 8) + threadIdx.x];
            }
        }
        __syncthreads();
    }

    // Final reduction and update global histogram
    if (threadIdx.x < 256) { 
        atomicAdd(&(d_hist[threadIdx.x]), aa[threadIdx.x]);
    }
}