#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogram_equalization_gpu_son (unsigned char * d_in, unsigned char * d_out, int * d_lut, int img_size, int serialNum)
{
    // Calculate global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if thread index is within image bounds
    if (x < img_size) {
        // Perform histogram equalization using lookup table
        d_out[x] = (unsigned char) d_lut[d_in[x]];
    }
}