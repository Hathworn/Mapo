#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned char clip_rgb_gpu(int x)
{
    if(x > 255) return 255;
    if(x < 0) return 0;
    return (unsigned char)x;
}

__global__ void yuv2rgb_gpu_son(unsigned char * d_y , unsigned char * d_u ,unsigned char * d_v , unsigned char * d_r, unsigned char * d_g, unsigned char * d_b, int size)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    if (x >= size) return;

    // Use local variables to reduce global memory access
    int y  = ((int)d_y[x]);
    int cb = ((int)d_u[x]) - 128;
    int cr = ((int)d_v[x]) - 128;

    // Pre-compute the common terms to optimize computation
    int cr_402 = 1.402 * cr;
    int cb_344 = 0.344 * cb;
    int cr_714 = 0.714 * cr;
    int cb_1772 = 1.772 * cb;

    int rt = y + cr_402;
    int gt = y - cb_344 - cr_714;
    int bt = y + cb_1772;

    d_r[x] = clip_rgb_gpu(rt);
    d_g[x] = clip_rgb_gpu(gt);
    d_b[x] = clip_rgb_gpu(bt);
}