#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clean(unsigned int * e, int n)
{
    // Use blockIdx and blockDim to handle larger arrays
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        e[idx] = 0; // Avoid modulo by using calculated index
    }
}