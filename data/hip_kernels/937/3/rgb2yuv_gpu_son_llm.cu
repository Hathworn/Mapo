#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rgb2yuv_gpu_son(unsigned char * d_r, unsigned char * d_g, unsigned char * d_b, unsigned char * d_y , unsigned char * d_u ,unsigned char * d_v , int size)
{
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    if (x >= size) return;
    
    // Utilize direct memory coalescing for increased access efficiency
    unsigned char r = d_r[x];
    unsigned char g = d_g[x];
    unsigned char b = d_b[x];

    // Precompute constant multiplications
    float y_coeff_r = 0.299f, y_coeff_g = 0.587f, y_coeff_b = 0.114f;
    float u_coeff_r = -0.169f, u_coeff_g = -0.331f, u_coeff_b = 0.499f;
    float v_coeff_r = 0.499f, v_coeff_g = -0.418f, v_coeff_b = -0.0813f;

    d_y[x] = (unsigned char)(y_coeff_r * r + y_coeff_g * g + y_coeff_b * b);
    d_u[x] = (unsigned char)(u_coeff_r * r + u_coeff_g * g + u_coeff_b * b + 128.0f);
    d_v[x] = (unsigned char)(v_coeff_r * r + v_coeff_g * g + v_coeff_b * b + 128.0f);
}