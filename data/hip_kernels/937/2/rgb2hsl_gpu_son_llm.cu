#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rgb2hsl_gpu_son(unsigned char *d_r, unsigned char *d_g, unsigned char *d_b, float *d_h, float *d_s, unsigned char *d_l, int size) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    if (x >= size) return;

    // Load RGB values and normalize to [0,1]
    float var_r = d_r[x] / 255.0f;
    float var_g = d_g[x] / 255.0f;
    float var_b = d_b[x] / 255.0f;
    
    // Calculate min and max RGB values
    float var_min = fminf(fminf(var_r, var_g), var_b);
    float var_max = fmaxf(fmaxf(var_r, var_g), var_b);
    float del_max = var_max - var_min;
    
    // Calculate luminance
    float L = (var_max + var_min) / 2.0f;

    float H = 0.0f, S = 0.0f;
    if (del_max != 0.0f) { // Chromatic (not gray)
        // Compute saturation
        S = (L < 0.5f) ? (del_max / (var_max + var_min)) : (del_max / (2.0f - var_max - var_min));

        // Compute hue
        float del_r = (((var_max - var_r) / 6.0f) + (del_max / 2.0f)) / del_max;
        float del_g = (((var_max - var_g) / 6.0f) + (del_max / 2.0f)) / del_max;
        float del_b = (((var_max - var_b) / 6.0f) + (del_max / 2.0f)) / del_max;
        if (var_r == var_max) H = del_b - del_g;
        else if (var_g == var_max) H = (1.0f / 3.0f) + del_r - del_b;
        else H = (2.0f / 3.0f) + del_g - del_r;

        if (H < 0.0f) H += 1.0f;
        if (H > 1.0f) H -= 1.0f;
    }

    // Store results back
    d_h[x] = H;
    d_s[x] = S;
    d_l[x] = static_cast<unsigned char>(L * 255.0f);
}