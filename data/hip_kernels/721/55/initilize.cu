#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initilize( unsigned int seed, hiprandState_t *states)
{
int tid = threadIdx.x + blockDim.x * blockIdx.x;
hiprand_init(
seed,
tid,
0,
&states[tid]);
}