#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initilize(unsigned int seed, hiprandState_t *states)
{
    // Optimize by computing tid only once
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialize random state for each thread
    hiprand_init(seed, tid, 0, &states[tid]);
}