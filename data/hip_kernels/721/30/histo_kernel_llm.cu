#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void histo_kernel(unsigned char *buffer1, long size1, unsigned int *histo1) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory for intermediate histogram
    __shared__ unsigned int local_histo[256];
    if (threadIdx.x < 256) local_histo[threadIdx.x] = 0;
    __syncthreads();

    // Each thread updates local histogram
    while (i < size1) {
        atomicAdd(&(local_histo[buffer1[i]]), 1);
        i += stride;
    }
    __syncthreads();

    // Accumulate local histograms into the global histogram
    if (threadIdx.x < 256) {
        atomicAdd(&(histo1[threadIdx.x]), local_histo[threadIdx.x]);
    }
}