#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addVectors(float *d_a, float *d_b, float *d_c, int size)
{
    // Cache size in shared memory to avoid repeated access to global memory
    __shared__ float shared_a[256];
    __shared__ float shared_b[256];

    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < size)
    {
        // Load data into shared memory for faster access
        shared_a[threadIdx.x] = d_a[i];
        shared_b[threadIdx.x] = d_b[i];
        __syncthreads(); // Ensure all threads have loaded their data

        // Perform vector addition using shared memory
        d_c[i] = shared_a[threadIdx.x] + shared_b[threadIdx.x];
    }
}