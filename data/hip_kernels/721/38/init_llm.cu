#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init( unsigned int seed, hiprandState_t *d_state)
{
    // Calculate global thread index once and reuse
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    hiprand_init(
        seed,
        idx,
        0,
        &d_state[idx]);
}