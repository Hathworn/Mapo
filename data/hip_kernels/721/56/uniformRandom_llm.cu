#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Load balance to improve warp efficiency by processing more elements per thread.
__global__ void uniformRandom(hiprandState_t *states, float *d_values) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int totalThreads = gridDim.x * blockDim.x;

    // Process multiple elements per thread to reduce kernel launch overhead
    for (int i = tid; i < N; i += totalThreads) {
        d_values[i] = hiprand_uniform(&states[i]);
    }
}