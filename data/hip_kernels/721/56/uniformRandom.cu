#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uniformRandom( hiprandState_t *states, float *d_values)
{
int tid = threadIdx.x + blockDim.x * blockIdx.x;
d_values[tid] = hiprand_uniform(&states[tid]);
}