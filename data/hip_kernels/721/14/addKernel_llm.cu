#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global index considering the block
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform addition if within bounds
    if (i < MAX_SIZE) {
        c[i] = a[i] + b[i];
    }
}
```
