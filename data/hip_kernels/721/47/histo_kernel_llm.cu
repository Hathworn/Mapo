#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel(unsigned char *buffer1, long size1, unsigned int *histo1){
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory to optimize atomic operations
    __shared__ unsigned int local_histo[256];
    if (threadIdx.x < 256) {
        local_histo[threadIdx.x] = 0;
    }
    __syncthreads();

    while (i < size1) {
        atomicAdd(&(local_histo[buffer1[i]]), 1);
        i += stride;
    }
    __syncthreads();

    // Write results from shared memory back to global memory
    if (threadIdx.x < 256) {
        atomicAdd(&(histo1[threadIdx.x]), local_histo[threadIdx.x]);
    }
}