#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void EFD(int size, float *d_val_n, float *d_val_npo, float Pu, float Pm, float Pd, float x0, float x)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < size)
    {
        float val_n_i = d_val_n[i];
        float val_n_plus1 = (i + 1 < size) ? d_val_n[i + 1] : 0; // Avoid out-of-bounds access
        float val_n_minus1 = (i > 0) ? d_val_n[i - 1] : 0; // Avoid out-of-bounds access
        d_val_npo[i] = Pu * val_n_plus1 + Pm * val_n_i + Pd * val_n_minus1; // Compute new value

        if (i == 0)
        {
            d_val_npo[i] = d_val_npo[1]; // Handle boundary condition at the start
        }
        else if (i == size - 1)
        {
            d_val_npo[i] = d_val_npo[i - 1]
                + exp(x0 + x * (float(i / 2)))
                - exp(x0 + x * (float(i / 2 - 1))); // Handle boundary condition at the end
        }
    }
}