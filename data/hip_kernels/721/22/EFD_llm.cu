#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void EFD(int size, float *d_val_n, float *d_val_npo, float Pu, float Pm, float Pd, float x0, float x)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < size)
    {
        // Performing boundary checks to prevent out-of-bounds access
        float val_n_plus = (i + 1 < size) ? d_val_n[i + 1] : 0.0f;
        float val_n_minus = (i - 1 >= 0) ? d_val_n[i - 1] : 0.0f;

        d_val_npo[i] = Pu * val_n_plus + Pm * d_val_n[i] + Pd * val_n_minus;

        if (i == 0)
        {
            d_val_npo[i] = d_val_npo[1];
        }
        else if (i == size - 1)
        {
            float exp_val1 = exp(x0 + x * float(i / 2));
            float exp_val2 = exp(x0 + x * float(i / 2 - 1));
            d_val_npo[i] = d_val_npo[i - 1] + exp_val1 - exp_val2;
        }
    }
}