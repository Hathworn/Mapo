#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float *x, int n)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride to cover all elements
    int stride = blockDim.x * gridDim.x;
    
    // Loop through the array with stride
    for (int i = tid; i < n; i += stride) {
        // Optimize calculation by avoiding repeated function calls
        float pi_pow_i = __powf(3.14159f, (float)i);
        x[i] = __fsqrt_rn(pi_pow_i);
    }
}