#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GeometricBrownianMotion(float *d_a, float mu, float sigma, float dt, hiprandState_t *states, unsigned int size)
{
    // Cache variables outside loop
    float sqrtDtSigma = sigma * sqrt(dt);
    float dtMu = dt * mu;

    // Optimize access and computation
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < size)
    {
        float randNormal = hiprand_normal(&states[i]);
        // Perform update using cached computations
        d_a[i] += d_a[i] * (dtMu + sqrtDtSigma * randNormal);
    }
}