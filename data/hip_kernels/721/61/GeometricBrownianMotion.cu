#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GeometricBrownianMotion( float *d_a, float mu, float sigma, float dt, hiprandState_t *states, unsigned int size)
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
if (i < size)
{
d_a[i] += d_a[i] * ( (dt*mu) + (sigma*sqrt(dt)*hiprand_normal(&states[i])));
}
}