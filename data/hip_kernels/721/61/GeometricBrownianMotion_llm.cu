#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GeometricBrownianMotion(float *d_a, float mu, float sigma, float dt, hiprandState_t *states, unsigned int size)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    // Use shared memory to reduce repeated sqrt calculation
    __shared__ float shared_sqrt_dt;

    // Only one thread calculates sqrt once, avoiding redundant computation
    if (threadIdx.x == 0) {
        shared_sqrt_dt = sqrt(dt);
    }
    __syncthreads();

    if (i < size)
    {
        // Using shared memory for sqrt(dt) to optimize performance
        d_a[i] += d_a[i] * ((dt * mu) + (sigma * shared_sqrt_dt * hiprand_normal(&states[i])));
    }
}