#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniDist(float *d_a, hiprandState_t *states, unsigned int size)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    // Use grid-stride loop to handle more elements per thread and improve occupancy
    for (; i < size; i += blockDim.x * gridDim.x)
    {
        d_a[i] = hiprand_uniform(&states[i]);
    }
}