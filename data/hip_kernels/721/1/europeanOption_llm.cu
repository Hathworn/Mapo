#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void europeanOption(int size, int iterations, float *d_price, float initialPrice, float strikePrice, hiprandState_t *d_state)
{
    // Calculate global thread index
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Ensure thread is within bounds
    if (tid < size)
    {
        float localPrice = initialPrice; // Use local variable for initialPrice

        // Pre-calculate constants outside the loop
        float mu_coef = mu / timespan;
        float sigma_coef = sigma / sqrt(timespan);

        for (int i = 0; i < iterations; i++)
        {
            // Optimize arithmetic operations inside the loop
            localPrice *= 1 + mu_coef + hiprand_normal(&d_state[tid]) * sigma_coef;
        }

        // Calculate option price and ensure non-negative value
        d_price[tid] = fmaxf(localPrice - strikePrice, 0.0f);
    }
}