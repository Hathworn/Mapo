#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uniDist( float *d_a, hiprandState_t *states, unsigned int size)
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
if (i < size)
{
d_a[i] = hiprand_uniform(&states[i]);
}
}