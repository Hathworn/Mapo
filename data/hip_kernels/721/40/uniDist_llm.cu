#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniDist(float* d_a, hiprandState_t* states, unsigned int size) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds; use local state copy for improved speed
    if (i < size) {
        hiprandState_t state = states[i];
        d_a[i] = hiprand_uniform(&state);
        states[i] = state; // Save state back to global memory
    }
}