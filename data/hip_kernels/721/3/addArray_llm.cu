#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addArray(float *d_a, float *d_b, float *d_c, int size) 
{
    // Calculate global index
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Ensure threads within bounds
    if (i < size) 
    {
        // Perform addition only if within bounds
        d_c[i] = d_a[i] + d_b[i];
    }
}