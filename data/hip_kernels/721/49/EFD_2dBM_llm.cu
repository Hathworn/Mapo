#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void EFD_2dBM(int width, int height, int pitch_n, int pitch_npo, float *d_val_n, float *d_val_npo, float alpha, float beta) {
    int idx = blockIdx.x;   // row
    int idy = threadIdx.x;  // column

    // Precompute commonly used terms to reduce redundant calculations
    int pitch_n_float = pitch_n / sizeof(float);
    int pitch_npo_float = pitch_npo / sizeof(float);

    if ((idx < height) && (idy < width)) {
        // Use shared memory for boundary handling (ghost cells)
        __shared__ float top, bottom, left, right, center;
        
        // Load values into shared variables
        center = d_val_n[idx * pitch_n_float + idy];
        top = (idx > 0) ? d_val_n[(idx - 1) * pitch_n_float + idy] : center;
        bottom = (idx < height - 1) ? d_val_n[(idx + 1) * pitch_n_float + idy] : center;
        left = (idy > 0) ? d_val_n[idx * pitch_n_float + idy - 1] : center;
        right = (idy < width - 1) ? d_val_n[idx * pitch_n_float + idy + 1] : center;

        // Compute the new value
        d_val_npo[idx * pitch_npo_float + idy] = alpha * (bottom + top)
                                                  + beta * (right + left)
                                                  + (1.0 - 2.0 * alpha - 2.0 * beta) * center;

        // Boundary conditions update
        if (idx == 0) {
            d_val_npo[idx * pitch_npo_float + idy] = d_val_npo[(idx + 1) * pitch_npo_float + idy];
        }
        if (idx == height - 1) {
            d_val_npo[idx * pitch_npo_float + idy] = d_val_npo[(idx - 1) * pitch_npo_float + idy];
        }
        if (idy == 0) {
            d_val_npo[idx * pitch_npo_float + idy] = d_val_npo[idx * pitch_npo_float + idy + 1];
        }
        if (idy == width - 1) {
            d_val_npo[idx * pitch_npo_float + idy] = d_val_npo[idx * pitch_npo_float + idy - 1];
        }
    }
}