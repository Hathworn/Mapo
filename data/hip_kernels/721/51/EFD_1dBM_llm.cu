#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void EFD_1dBM(int size, float *d_val_n, float *d_val_npo, float Pu, float Pm, float Pd) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < size) {
        // Improved memory coalescing
        float val_npo = 0.0f;
        if (i > 0) {
            val_npo += Pd * d_val_n[i - 1];
        }
        val_npo += Pm * d_val_n[i];
        if (i < size - 1) {
            val_npo += Pu * d_val_n[i + 1];
        }

        d_val_npo[i] = val_npo;

        // Simplified boundary handling with conditional avoidance
        if (i == 0) {
            d_val_npo[i] = d_val_npo[1];
        } else if (i == size - 1) {
            d_val_npo[i] = d_val_npo[i - 1];
        }
    }
}