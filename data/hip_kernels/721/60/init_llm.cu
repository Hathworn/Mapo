#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(unsigned int seed, hiprandState_t *states, unsigned int size)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (i < size)
    {
        // Initialize random state
        hiprand_init(seed, i, 0, &states[i]);
    }
}