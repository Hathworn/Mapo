#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init( unsigned int seed, hiprandState_t *states, unsigned int size)
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
if (i < size)
{
hiprand_init(
seed,
i,
0,
&states[i]);
}
}