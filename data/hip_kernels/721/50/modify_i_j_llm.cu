#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void modify_i_j(int width, int height, int pitch, float *d_array, int i, int j, float change_to) {
    // Calculate row and column using blockIdx and threadIdx directly
    int idx = blockIdx.x;  // row
    int idy = threadIdx.x; // column

    // Directly access and modify the element if conditions are met
    if (idx == i && idy == j) {
        d_array[idx * (pitch / sizeof(float)) + idy] = change_to;
    }
}