#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel(unsigned char *buffer1, long size1, unsigned int *histo1) {

    // Phase 1 ------------------------------------------------------------
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use loop unrolling to optimize memory access within the while loop
    while (i < size1) {
        atomicAdd(&(temp[buffer1[i]]), 1);
        if (i + blockDim.x < size1) atomicAdd(&(temp[buffer1[i + blockDim.x]]), 1);
        if (i + 2 * blockDim.x < size1) atomicAdd(&(temp[buffer1[i + 2 * blockDim.x]]), 1);
        if (i + 3 * blockDim.x < size1) atomicAdd(&(temp[buffer1[i + 3 * blockDim.x]]), 1);
        i += 4 * stride;  // Adjust stride for loop unrolling
    }
    __syncthreads();
    //---------------------------------------------------------------------

    // Phase 2 ------------------------------------------------------------
    atomicAdd(&(histo1[threadIdx.x]), temp[threadIdx.x]);
    //---------------------------------------------------------------------
}