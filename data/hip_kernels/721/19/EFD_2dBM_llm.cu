#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void EFD_2dBM(int width, int height, int pitch_n, int pitch_npo, float* d_val_n, float* d_val_npo, float alpha, float beta) {
    int idx = blockIdx.x; // row
    int idy = threadIdx.x; // column

    if ((idx < height) && (idy < width)) {
        // Pre-compute common indices
        int idx_pitch_n = idx * (pitch_n / sizeof(float));
        int idx_pitch_npo = idx * (pitch_npo / sizeof(float));

        // Load neighboring values
        float north = (idx > 0) ? d_val_n[(idx - 1) * (pitch_n / sizeof(float)) + idy] : d_val_n[1 * (pitch_n / sizeof(float)) + idy];
        float south = (idx < height - 1) ? d_val_n[(idx + 1) * (pitch_n / sizeof(float)) + idy] : d_val_n[(height - 2) * (pitch_n / sizeof(float)) + idy];
        float west = (idy > 0) ? d_val_n[idx_pitch_n + idy - 1] : d_val_n[idx_pitch_n + 1];
        float east = (idy < width - 1) ? d_val_n[idx_pitch_n + idy + 1] : d_val_n[idx_pitch_n + width - 2];

        // Perform stencil operation
        d_val_npo[idx_pitch_npo + idy] = alpha * (north + south) + beta * (west + east) + (1.0f - 2.0f * alpha - 2.0f * beta) * d_val_n[idx_pitch_n + idy];
    }
}