#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *x, int n)
{
    // Calculate global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Use stride loop with precomputed constant value
    const float constant = sqrt(3.14159);
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        x[i] = powf(constant, i); // Use powf for single precision
    }
}