#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(unsigned int seed, hiprandState_t *states, unsigned int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size)
    {
        // Utilize registers for seed calculation
        hiprandState_t localState;
        hiprand_init(seed, i, 0, &localState);
        states[i] = localState;
    }
}