#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void europeanOption( int size, int iterations, float *d_price, float initialPrice, float strikePrice, hiprandState_t *d_state)
{
int tid = threadIdx.x + blockDim.x * blockIdx.x;

if (tid < size)
{

for (int i = 0; i < iterations; i++)
{
initialPrice *= 1 + mu / timespan + hiprand_normal(&d_state[tid])*sigma/sqrt(timespan);
}

d_price[tid] = initialPrice - strikePrice;
if (d_price[tid] < 0)
{
d_price[tid] = 0;
}
}

}