#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void europeanOption(int size, int iterations, float *d_price, float initialPrice, float strikePrice, hiprandState_t *d_state)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < size)
    {
        float localPrice = initialPrice; // Move variable for thread-local computation

        // Use a local variable to avoid repeated calculations
        float mu_timespan_factor = mu / timespan;
        float sigma_sqrt_timespan = sigma / sqrt(timespan);

        for (int i = 0; i < iterations; i++)
        {
            localPrice *= 1 + mu_timespan_factor + hiprand_normal(&d_state[tid]) * sigma_sqrt_timespan;
        }

        // Use a single calculation and store only needed data
        d_price[tid] = fmaxf(localPrice - strikePrice, 0.0f);
    }
}