#include "hip/hip_runtime.h"
```cuda
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation
    if (i < N)  // Prevent out-of-bounds memory access (Assuming N is defined and known)
    {
        c[i] = a[i] + b[i]; // Perform vector addition
    }
}