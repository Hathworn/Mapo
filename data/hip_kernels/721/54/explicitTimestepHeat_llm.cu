#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float explicitLocalStepHeat(float unjpo, float unjmo, float unj, float r) {
    return (1 - 2 * r) * unj + r * unjmo + r * unjpo;
}

__global__ void explicitTimestepHeat(int size, float *d_currentVal, float *d_nextVal, float r) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Optimized indexing for clarity and standard convention
    if (i < size) {
        if (i < 2 || i > size - 2) {  // Combined boundary checks for clarity and efficiency
            d_nextVal[i] = 0;  // Fixed assignment operator from '==' to '='
        } else {
            d_nextVal[i] = explicitLocalStepHeat(
                d_currentVal[i + 1],
                d_currentVal[i - 1],
                d_currentVal[i],
                r
            );
        }
    }
}