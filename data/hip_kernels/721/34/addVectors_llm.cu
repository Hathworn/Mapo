#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addVectors(int size, float *d_a, float *d_b, float *d_c)
{
    // Precalculate thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce global memory access latency
    extern __shared__ float cache[];
    
    if (tid < size)
    {
        cache[threadIdx.x] = d_a[tid] + d_b[tid]; // Load into shared memory
        __syncthreads();  // Ensure all writes to shared memory are completed

        d_c[tid] = cache[threadIdx.x]; // Write back to global memory
    }
}