#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void modify_i_j(int width, int height, int pitch, float *d_array, int i, int j, float change_to) {
    // Utilizing block and thread for efficient indexing
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate row index
    int idy = blockIdx.y * blockDim.y + threadIdx.y; // Calculate column index

    // Ensure we are within the array bounds
    if (idx < width && idy < height) {
        // More efficient indexing and only modify the desired element
        if ((idx == i) && (idy == j)) {
            d_array[idx * (pitch / sizeof(float)) + idy] = change_to;
        }
    }
}