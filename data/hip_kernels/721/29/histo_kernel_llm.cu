#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel(unsigned char *buffer1, long size1, unsigned int *histo1) {

    // Phase 1: Use shared memory for temporary histogram
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Process each stride, optimizing loop with local reduction
    while (i < size1) {
        atomicAdd(&(temp[buffer1[i]]), 1);
        i += stride;
    }
    __syncthreads();

    // Phase 2: Accumulate local histogram into global histogram
    atomicAdd(&(histo1[threadIdx.x]), temp[threadIdx.x]);
}