#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addArray(float *d_a, float *d_b, float *d_c, int size)
{
    // Use shared memory for caching inputs
    extern __shared__ float shared_a[];
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int t = threadIdx.x;

    if (i < size)
    {
        // Load inputs into shared memory
        shared_a[t] = d_a[i];
        __syncthreads();

        if (i < size)
        {
            // Efficient array addition using shared memory
            d_c[i] = shared_a[t] + d_b[i];
        }
    }
}