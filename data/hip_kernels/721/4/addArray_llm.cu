#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel
__global__ void addArray(float *d_a, float *d_b, float *d_c, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Avoid thread divergence by using loop
    while (i < size) {
        d_c[i] = d_a[i] + d_b[i];
        i += blockDim.x * gridDim.x; // Move to next relevant index
    }
}