#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void init( unsigned int seed, hiprandState_t *d_state)
{
    // Calculate thread global index once for efficiency
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    hiprand_init(seed, idx, 0, &d_state[idx]);
}