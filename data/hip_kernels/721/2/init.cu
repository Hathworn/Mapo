#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init( unsigned int seed, hiprandState_t *d_state)
{
hiprand_init(
seed,
threadIdx.x + blockDim.x * blockIdx.x,
0,
&d_state[threadIdx.x + blockDim.x * blockIdx.x]);
}