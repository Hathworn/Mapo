#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void curandINIT(int size, hiprandState_t *d_states, unsigned int seed)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop to ensure all threads can contribute when grid is smaller than size
    for (int idx = i; idx < size; idx += blockDim.x * gridDim.x)
    {
        // Initialize state with a unique seed
        hiprand_init(seed, idx, 0, &d_states[idx]);
    }
}