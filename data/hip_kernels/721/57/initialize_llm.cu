#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initialize( unsigned int seed, hiprandState_t *states, unsigned int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) 
    {
        // Initialize the RNG state directly with the index
        hiprand_init(seed, i, 0, &states[i]);
    }
}