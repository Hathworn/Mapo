#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void initialize( unsigned int seed, hiprandState_t *states, unsigned int size)
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
if (i < size)
{
hiprand_init(
seed,
i,
0,
&states[i]);
}
}