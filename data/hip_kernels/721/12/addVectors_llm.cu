#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addVectors(float *d_A, float *d_B, float *d_C, int size)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // Use a loop to process multiple elements per thread to optimize memory access
    int stride = blockDim.x * gridDim.x;
    for (; i < size; i += stride)
    {
        d_C[i] = d_A[i] + d_B[i];
    }
}