#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void EFD_1dBM(int size, float *d_val_n, float *d_val_npo, float Pu, float Pm, float Pd) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < size) {
        float new_val_i;
        if (i > 0 && i < size - 1) {
            new_val_i = Pu * d_val_n[i + 1] + Pm * d_val_n[i] + Pd * d_val_n[i - 1];
        } else {
            new_val_i = Pm * d_val_n[i];  // Boundary element calculation
        }

        if (i == 0) {
            d_val_npo[i] = d_val_npo[1];
        } else if (i == size - 1) {
            d_val_npo[i] = d_val_npo[i - 1];
        } else {
            d_val_npo[i] = new_val_i;  // Store calculated value
        }
    }
}