#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void curandINIT(int size, hiprandState_t *d_states, unsigned int seed)
{
    // Utilize a single calculation to improve performance
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure threads do not go out of bounds
    if (i < size)
    {
        // Initialize the random state more efficiently
        hiprand_init(seed, i, 0, &d_states[i]);
    }
}