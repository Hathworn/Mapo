#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void curandINIT( int size, hiprandState_t *d_states, unsigned int seed)
{
int i = threadIdx.x + blockDim.x * blockIdx.x;
if (i < size)
{
hiprand_init(seed, i, 0, &d_states[i]);
}
}