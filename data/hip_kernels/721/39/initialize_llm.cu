#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initialize(unsigned int seed, hiprandState_t *states, unsigned int size)
{
    // Calculate the global index once for better performance
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread does not exceed array bounds
    if (i < size)
    {
        // Initialize the random state for each thread
        hiprand_init(seed, i, 0, &states[i]);
    }
}