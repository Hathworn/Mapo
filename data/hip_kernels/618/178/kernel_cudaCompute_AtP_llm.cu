#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_cudaCompute_AtP(double *d_A, double *d_P, double *d_AtP, int rows, int columns)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = rows * columns;

    // Unrolling loop for performance optimization
    for (int i = tid; i < total_elements; i += blockDim.x * gridDim.x)
    {
        int row = i % rows;
        int column = i / rows;
        d_AtP[row + column * rows] = d_A[column + row * columns] * d_P[column];
    }
}