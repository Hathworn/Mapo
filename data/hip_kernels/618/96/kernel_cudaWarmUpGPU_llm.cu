#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use a kernel that does actual work for warmup
__global__ void kernel_cudaWarmUpGPU()
{
    // Calculate global thread index
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    // Simple dummy computation to keep GPU busy
    if (threadId % 2 == 0) {
        threadId *= threadId;
    } else {
        threadId += threadId;
    }
}