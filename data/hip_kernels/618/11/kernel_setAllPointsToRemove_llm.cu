#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_setAllPointsToRemove(bool *d_markers, int number_of_points)
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind < number_of_points)
    {
        // Utilize coalesced memory access
        d_markers[ind] = false;
    }
}