#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_cudaPrepareProjectionIndexes(char *d_v_is_projection, int *d_nearest_neighbour_indexes, int number_of_points)
{
    // Calculate global thread index
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure valid index for processing
    if (ind < number_of_points)
    {
        // Use a single-line conditional to update nearest neighbour indexes
        d_nearest_neighbour_indexes[ind] = (d_v_is_projection[ind] == 0) ? -1 : ind;
    }
}