#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function for computing AtP
__global__ void kernel_cudaCompute_AtP(double *d_A, double *d_P, double *d_AtP, int rows, int columns)
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use if condition to avoid out-of-bound memory access
    if (ind < rows * columns) 
    {
        // Pre-compute indices to reduce repetitive calculations
        int column = ind / rows; 
        int row = ind % rows;

        // Perform computation using pre-computed indices
        d_AtP[row + column * rows] = d_A[column + row * columns] * d_P[column];
    }
}