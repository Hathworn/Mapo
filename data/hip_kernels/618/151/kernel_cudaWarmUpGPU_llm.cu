#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_cudaWarmUpGPU()
{
    // Optimize thread index calculation
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    // Use ind for some simple operation to avoid compiler optimization
    ind = __syncthreads_count(ind) + 1;
}