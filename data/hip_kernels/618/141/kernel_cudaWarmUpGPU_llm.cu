#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_cudaWarmUpGPU()
{
    // Compute the global index
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure all threads reach this point
    __syncthreads();
}