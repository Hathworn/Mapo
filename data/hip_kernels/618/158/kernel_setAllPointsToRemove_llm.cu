#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define a constant for better readability and code maintenance
#define THREADS_PER_BLOCK 256 

__global__ void kernel_setAllPointsToRemove(int number_of_points, bool *d_markers_out)
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    // Use strided loop for better GPU utilization
    for (int i = ind; i < number_of_points; i += blockDim.x * gridDim.x)
    {
        d_markers_out[i] = false;
    }
}