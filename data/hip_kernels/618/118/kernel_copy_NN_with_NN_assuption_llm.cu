#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_copy_NN_with_NN_assumption(double *d_temp_double_mem, int *d_nearest_neighbour_indexes, int number_of_points)
{
    // Calculate the global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure threads within valid range only perform computation
    if (index < number_of_points)
    {
        int i = d_nearest_neighbour_indexes[index];

        // Use ternary operator for concise condition assignment
        d_temp_double_mem[index] = (i != -1) ? 1.0 : 0.0;
    }
}