#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_cudaWarmUpGPU()
{
    // Calculate unique thread index more efficiently
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind < blockDim.x * gridDim.x) {
        // Perform a simple operation to warm up the GPU
        ind++; // This line just to avoid compiler optimizations
    }
}