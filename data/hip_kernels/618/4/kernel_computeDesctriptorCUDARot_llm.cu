#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 26
#define RADIUS 3
#define BLOCK_SIZE (TILE_SIZE+(2*RADIUS))

texture<unsigned char, 1, hipReadModeElementType> texInImage;
texture<unsigned int, 1, hipReadModeElementType> texIntegralImage;

__device__ unsigned int keypointsCount = 0;

__global__ void kernel_computeDesctriptorCUDARot(bool *_d_isdescriptor, char *_d_vdescriptor, int *_d_keypointsIndexX, int *_d_keypointsIndexY, int *_d_keypointsRotation, int _amountofkeypoints, unsigned int *_d_integralImage, int _d_width, int _d_height, float _scale)
{
    int bx = blockIdx.x;
    int tx = threadIdx.x;

    int index = bx + tx * _d_height;
    _d_isdescriptor[index] = false;

    // Early exit for out-of-bounds index
    if (index >= _amountofkeypoints) return;

    float S[64];
    float _X[64];
    float _Y[64];
    float _Z[64];
    float r, phi;
    const float pi = 3.1415926535f;
    
    // Precompute values for _X, _Y, _Z
    for (int i = 0; i < 64; i++)
    {
        r = _scale * pow(2.0f, 2 + (i % 4));
        phi = (float)(i) / 4.0f;
        _X[i] = r * cos((2.0f * pi * phi) / 16.0f);
        _Y[i] = r * sin((2.0f * pi * phi) / 16.0f);
        _Z[i] = _scale * 8;
    }

    int _xIndex = _d_keypointsIndexX[index];
    int _yIndex = _d_keypointsIndexY[index];
    int tau = 4 * _d_keypointsRotation[index];

    bool check = true;
    int index0, index1, index2, index3;

    // Prefetch width and height
    int _h_width = _d_width;
    int _h_height = _d_height;
    
    // Consolidate index checks within loop
    for (int i = 0; i < 64; i++)
    {
        int xOffsetPos = floor(_X[i]) + floor(_Z[i]);
        int xOffsetNeg = floor(_X[i]) - floor(_Z[i]);
        int yOffsetPos = floor(_Y[i]) + floor(_Z[i]);
        int yOffsetNeg = floor(_Y[i]) - floor(_Z[i]);
        
        int idxPosPos = _xIndex + xOffsetPos + (_yIndex + yOffsetPos) * _h_width;
        int idxNegNeg = _xIndex + xOffsetNeg + (_yIndex + yOffsetNeg) * _h_width;
        int idxPosNeg = _xIndex + xOffsetPos + (_yIndex + yOffsetNeg) * _h_width;
        int idxNegPos = _xIndex + xOffsetNeg + (_yIndex + yOffsetPos) * _h_width;
        
        if (idxPosPos < 0 || idxPosPos >= _h_width * _h_height ||
            idxNegNeg < 0 || idxNegNeg >= _h_width * _h_height ||
            idxPosNeg < 0 || idxPosNeg >= _h_width * _h_height ||
            idxNegPos < 0 || idxNegPos >= _h_width * _h_height) {
            check = false;
            break;
        }

        index0 = idxPosPos;
        index1 = idxNegNeg;
        index2 = idxPosNeg;
        index3 = idxNegPos;

        unsigned int a1 = tex1Dfetch(texIntegralImage, index0);
        unsigned int a2 = tex1Dfetch(texIntegralImage, index1);
        unsigned int a3 = tex1Dfetch(texIntegralImage, index2);
        unsigned int a4 = tex1Dfetch(texIntegralImage, index3);

        S[i] = float(a1 + a2 - a3 - a4);
    }

    if (check)
    {
        _d_isdescriptor[index] = true;
        bool desc[256];

        // Optimize descriptor computation
        for (int i = 0; i < 64; i++)
        {
            int id = (i + tau) % 64;
            int index0 = (id + 8) % 64;
            int index1 = (id + 24) % 64;
            int index2 = (id + 36) % 64;
            int index3 = int((4.0f * id / 4.0f + 4.0f + (3 - (id % 4)))) % 64;

            desc[i * 4] = S[id] < S[index0];
            desc[i * 4 + 1] = S[id] < S[index1];
            desc[i * 4 + 2] = S[id] < S[index2];
            desc[i * 4 + 3] = S[id] < S[index3];
        }

        // Pack boolean array into char descriptors
        for (int i = 0; i < 32; i++)
        {
            char wynik = 0;
            for (int j = 0; j < 8; j++)
            {
                wynik |= (desc[i * 8 + j] << j);
            }
            _d_vdescriptor[index * 32 + i] = wynik;
        }
    }
}