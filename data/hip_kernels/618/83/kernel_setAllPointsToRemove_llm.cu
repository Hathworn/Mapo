#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_setAllPointsToRemove(bool *d_markers, int number_of_points)
{
    // Calculate global thread index
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Improved bounds check to eliminate redundant thread execution
    if (ind < number_of_points)
    {
        d_markers[ind] = false; // Directly set marker to false
    }
}