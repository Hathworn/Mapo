#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 26
#define RADIUS 3
#define BLOCK_SIZE (TILE_SIZE+(2*RADIUS))

texture<unsigned char, 1, hipReadModeElementType> texInImage;
texture<unsigned int, 1, hipReadModeElementType> texIntegralImage;

__device__ unsigned int keypointsCount = 0;

__global__ void kernel_initIntegrtalImage(unsigned int *_d_out_integralImage, int _h_width, int _h_height)
{
    // Calculate 1D global thread ID
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check boundary before assignment
    if (index < _h_width * _h_height) {
        _d_out_integralImage[index] = 0;
    }
}