#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel for CUDA warm-up
__global__ void kernel_cudaWarmUpGPU()
{
    // Calculate global thread index
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    // Use the index for warm-up computation to ensure it doesn't get optimized away
    if (ind < 1) // Single-thread execution for validation
    {
        // Volatile variable to prevent optimization
        volatile int warmUp = ind + 1;
    }
}