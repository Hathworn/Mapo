#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to warm up the GPU
__global__ void kernel_cudaWarmUpGPU()
{
    // Calculate global thread index
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform a no-op calculation to prevent compiler optimizations
    if (ind) {
        ind = ind + 1;
    }
}