#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_cudaWarmUpGPU()
{
    // Calculate unique global thread index
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use the index to prevent compiler optimization
    if (ind < 0) {
        printf("%d\n", ind);
    }
}