#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_copy_NN_with_NN_assuption(double *d_temp_double_mem, int *d_nearest_neighbour_indexes, int number_of_points)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Load d_nearest_neighbour_indexes[index] into a register to reduce repeated global memory access
    if (index < number_of_points) {
        int i = d_nearest_neighbour_indexes[index];
        d_temp_double_mem[index] = (i != -1) ? 1.0f : 0.0f; // Use a ternary operator for simplicity
    }
}