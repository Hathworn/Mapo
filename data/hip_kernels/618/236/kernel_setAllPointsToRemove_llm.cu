#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_setAllPointsToRemove(bool *d_markers, int number_of_points)
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimization: Use stride-based loop to handle more elements
    for (int i = ind; i < number_of_points; i += blockDim.x * gridDim.x)
    {
        d_markers[i] = false;
    }
}