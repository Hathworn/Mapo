#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_fill_nn_cuda(unsigned int *d_nn, int *nearest_neighbour_indexes, unsigned int number_nearest_neighbour_indexes)
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    // Check within bounds to avoid branching inside conditional
    if (ind < number_nearest_neighbour_indexes)
    {
        d_nn[ind] = (nearest_neighbour_indexes[ind] < 0) ? 0 : 1;  // Use ternary operator for conciseness
    }
}