#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_copy_NN_with_NN_assuption(double *d_temp_double_mem, int *d_nearest_neighbour_indexes, int number_of_points)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use branchless computing for better performance
    if(index < number_of_points)
    {
        int i = d_nearest_neighbour_indexes[index];
        d_temp_double_mem[index] = (i != -1) ? 1.0 : 0.0;
    }
}