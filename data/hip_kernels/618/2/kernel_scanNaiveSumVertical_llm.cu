#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 26
#define RADIUS 3
#define BLOCK_SIZE (TILE_SIZE+(2*RADIUS))

texture<unsigned char, 1, hipReadModeElementType> texInImage;
texture<unsigned int, 1, hipReadModeElementType> texIntegralImage;

__device__ unsigned int keypointsCount = 0;

__global__ void kernel_scanNaiveSumVertical(unsigned int *_d_out_integralImage, unsigned char *_d_in_image, int _h_width, int _h_height)
{
    // Utilize shared memory for fast access
    __shared__ unsigned int tmp[TILE_SIZE];

    int tx = threadIdx.x;
    int index = tx * _h_width;
    
    // Load elements into shared memory
    for(int i = 0 ; i < _h_width; i++)
    {
        tmp[i] = (unsigned int)tex1Dfetch(texInImage, index + i);
    }
    
    __syncthreads();  // Ensure shared memory is populated

    // Perform the scan operation
    for(int i = 1; i < _h_width; i++)
    {
        tmp[i] = tmp[i - 1] + tmp[i];
    }
    
    __syncthreads();  // Ensure scan is completed
    
    // Write back to global memory
    for(int i = 0 ; i < _h_width; i++)
    {
        _d_out_integralImage[index + i] = tmp[i];
    }
}