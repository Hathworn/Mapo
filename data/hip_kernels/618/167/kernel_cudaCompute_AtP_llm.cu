#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_cudaCompute_AtP(double *d_A, double *d_P, double *d_AtP, int rows, int columns )
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit early if index is out of bounds
    if (ind >= rows * columns) return;

    int row = ind % rows;
    int column = ind / rows;

    // Use shared memory for A and P values to improve access speed when possible
    __shared__ double shared_A[columns];
    __shared__ double shared_P[columns];

    shared_A[column] = d_A[column + row * columns];
    shared_P[column] = d_P[column];

    // Ensures all threads in the block have reached this point before proceeding
    __syncthreads();

    // Compute the resulting d_AtP value with the shared memory
    d_AtP[row + column * rows] = shared_A[column] * shared_P[column];
}