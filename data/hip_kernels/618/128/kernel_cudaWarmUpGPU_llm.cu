#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_cudaWarmUpGPU()
{
    // Use volatile to prevent compiler optimization from removing the operation
    volatile int ind = blockIdx.x * blockDim.x + threadIdx.x;
    ind = ind + 1; // Simple increment operation to engage the GPU
}