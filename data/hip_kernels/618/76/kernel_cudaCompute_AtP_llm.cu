#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_cudaCompute_AtP(double *d_A, double *d_P, double *d_AtP, int rows, int columns) {
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for faster access to frequently used data
    __shared__ double shared_P[256]; // Adjust size based on blockDim.x

    if (threadIdx.x < columns) {
        shared_P[threadIdx.x] = d_P[threadIdx.x];
    }
    __syncthreads();

    if (ind < rows * columns) {
        int row = ind % rows;
        int column = ind / rows;

        // Access shared memory for P to reduce global memory access latency
        d_AtP[row + column * rows] = d_A[column + row * columns] * shared_P[column];
    }
}