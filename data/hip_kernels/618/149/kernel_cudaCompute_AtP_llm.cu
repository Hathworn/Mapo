#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_cudaCompute_AtP(double *d_A, double *d_P, double *d_AtP, int rows, int columns)
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = rows * columns;
    
    // Ensure each thread processes multiple elements for better utilization
    for (int i = ind; i < total_elements; i += gridDim.x * blockDim.x)
    {
        int row = i % rows;
        int column = i / rows;
        
        // Pre-compute memory access indices for coalesced memory access
        int a_index = column + row * columns;
        int atp_index = row + column * rows;
        
        d_AtP[atp_index] = d_A[a_index] * d_P[column];
    }
}