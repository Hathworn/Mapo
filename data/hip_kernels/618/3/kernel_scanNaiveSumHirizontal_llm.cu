#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 26
#define RADIUS 3
#define BLOCK_SIZE (TILE_SIZE+(2*RADIUS))

texture<unsigned char, 1, hipReadModeElementType> texInImage;
texture<unsigned int, 1, hipReadModeElementType> texIntegralImage;

__device__ unsigned int keypointsCount = 0;

// Optimized kernel function to perform prefix sum horizontally
__global__ void kernel_scanNaiveSumHirizontal(unsigned int *_d_out_integralImage, int _h_width, int _h_height)
{
    extern __shared__ unsigned int s_data[];  // Use shared memory
    int tx = threadIdx.x;
    int gx = blockIdx.x * blockDim.x + tx;

    if (gx < _h_width) {
        // Load data from texture to shared memory
        for (int i = 0; i < _h_height; i++) {
            s_data[i * blockDim.x + tx] = tex1Dfetch(texIntegralImage, gx + i * _h_width);
        }

        // Perform scan operation in shared memory
        for (int i = 1; i < _h_height; i++) {
            s_data[i * blockDim.x + tx] += s_data[(i - 1) * blockDim.x + tx];
        }
        
        // Write results back to global memory
        for (int i = 0; i < _h_height; i++) {
            _d_out_integralImage[gx + i * _h_width] = s_data[i * blockDim.x + tx];
        }
    }
}