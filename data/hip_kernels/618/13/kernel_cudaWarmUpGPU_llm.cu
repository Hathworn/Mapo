#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_cudaWarmUpGPU()
{
    // Calculate global thread index
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform a no-op to warm up GPU
    if (ind < 1) // Optimize with a condition, though here it's a warm-up example
    {
        ind += 1;
    }
}