#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_cudaWarmUpGPU()
{
    // Calculate unique global thread index
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    // The operation below is a no-op, remove as it doesn't serve any function.
    // ind = ind + 1;

    // To effectively "warm-up," ensure all threads are active by executing a dummy operation
    if (ind >= 0) {
        __syncthreads();  // Synchronize to ensure all threads have reached this point
    }
}