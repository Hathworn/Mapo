#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_cudaWarmUpGPU()
{
    // Calculate global thread index
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    // No operation needed but ensure the thread is being used to prevent optimization
    asm volatile(""); // Compiler barrier to prevent removal of ind
}