#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_cudaWarmUpGPU()
{
    // Calculate global thread index
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop to utilize and keep GPU active
    for (int i = 0; i < 1000; i++) {
        ind = ind + 1; // Simple operation to keep activity
    }
}