#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void	randInitGpu (hiprandState_t * state, const uint seed, const uint rank, const uint size)
{
uint bIdx = blockIdx.x + gridDim.x*blockIdx.y;
uint idx  = threadIdx.x + blockDim.x*bIdx;

hiprand_init (seed*gridDim.x*gridDim.y + rank*size*gridDim.x*gridDim.y + bIdx, threadIdx.x, 0, &state[idx]);
}