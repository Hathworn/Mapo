#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randInitGpu(hiprandState_t *state, const uint seed, const uint rank, const uint size) {
    uint bIdx = blockIdx.x + gridDim.x * blockIdx.y;
    uint idx = threadIdx.x + blockDim.x * bIdx;

    // Use a single expression to calculate initialization parameter
    uint init_param = seed * gridDim.x * gridDim.y + rank * size * gridDim.x * gridDim.y + bIdx;
    
    // Initialize state using calculated parameter
    hiprand_init(init_param, idx, 0, &state[idx]);
}