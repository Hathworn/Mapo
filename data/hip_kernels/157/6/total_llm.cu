#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 256

__global__ void total(float * input, float * output, int len) {
    // Allocate shared memory dynamically
    extern __shared__ float partialSum[];

    unsigned int tx = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * BLOCK_SIZE;

    // Initialize shared memory with input values or zero if out of bounds
    if ((start + tx) < len) {
        partialSum[tx] = input[start + tx];
    } else {
        partialSum[tx] = 0.0;
    }
    if ((start + BLOCK_SIZE + tx) < len) {
        partialSum[BLOCK_SIZE + tx] = input[start + BLOCK_SIZE + tx];
    } else {
        partialSum[BLOCK_SIZE + tx] = 0.0;
    }

    // Reduce the values in shared memory
    for (unsigned int stride = BLOCK_SIZE; stride > 0; stride /= 2) {
        __syncthreads();
        if (tx < stride) {
            partialSum[tx] += partialSum[tx + stride];
        }
    }

    // Store the result in the output array
    if (tx == 0) {
        output[blockIdx.x] = partialSum[0];
    }
}