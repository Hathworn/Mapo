#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAddKernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) // Boundary check for threads.
    {
        C[i] = A[i] + B[i]; // Use of __restrict__ to improve memory access
    }
}