#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define HISTOGRAM_LENGTH 256

__global__ void histo_kernel(unsigned char *buffer, unsigned int *histo, long size) {
    __shared__ unsigned int histo_private[HISTOGRAM_LENGTH];

    int tx = threadIdx.x;

    // Initialize shared histogram
    if (tx < HISTOGRAM_LENGTH) 
        histo_private[tx] = 0;

    __syncthreads();

    // Use a stride loop to distribute read tasks among threads
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    while (index < size) {
        atomicAdd(&(histo_private[buffer[index]]), 1);
        index += stride;  // Efficient stride-based iteration for full buffer access
    }

    __syncthreads();

    // Merge shared histogram back to global histogram
    if (tx < HISTOGRAM_LENGTH) 
        atomicAdd(&(histo[tx]), histo_private[tx]);
}