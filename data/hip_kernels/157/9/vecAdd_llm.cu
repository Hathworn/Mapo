#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform vector addition if index is within bounds
    if (idx < len) {
        out[idx] = in1[idx] + in2[idx];
    }
}