#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan(float * input, float * output, int len) {
    // Load into shared memory or zero if out of bounds
    __shared__ float scan_array[BLOCK_SIZE];
    int global_id = threadIdx.x + blockDim.x * blockIdx.x;
    scan_array[threadIdx.x] = (global_id < len) ? input[global_id] : 0;
    __syncthreads();

    // Up-sweep (reduce) phase
    for (int stride = 1; stride < BLOCK_SIZE; stride *= 2) {
        int index = (threadIdx.x + 1) * 2 * stride - 1;
        if (index < BLOCK_SIZE) scan_array[index] += scan_array[index - stride];
        __syncthreads();
    }

    // Down-sweep phase
    for (int stride = BLOCK_SIZE >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        int index = (threadIdx.x + 1) * 2 * stride - 1;
        if (index + stride < BLOCK_SIZE) scan_array[index + stride] += scan_array[index];
    }
    __syncthreads();

    // Write results to global memory
    if (global_id < len) output[global_id] = scan_array[threadIdx.x];

    // Handle boundary condition and carry over sums to next blocks
    if (threadIdx.x == 0) {
        float block_sum = scan_array[BLOCK_SIZE - 1];
        for (int block_idx = blockIdx.x + 1; block_idx < (len + BLOCK_SIZE - 1) / BLOCK_SIZE; ++block_idx) {
            int offset_global_id = block_idx * BLOCK_SIZE + threadIdx.x;
            if (offset_global_id < len) output[offset_global_id] += block_sum;
            __syncthreads();
        }
    }
}