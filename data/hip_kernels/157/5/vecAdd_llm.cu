#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
    // Calculate global thread ID and ensure within bounds
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len) {
        out[i] = in1[i] + in2[i]; // Vector addition
    }
}