#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define HISTOGRAM_LENGTH 256

__global__ void convertToChar(float *input, unsigned char *ucharInput, int width, int height)
{
    // Calculate global thread position
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check boundaries
    if (row < height && col < width)
    {
        int index3 = (row * width + col) * 3;

        // Convert float to unsigned char
        ucharInput[index3]     = __saturatef(input[index3]) * 255;   // r
        ucharInput[index3 + 1] = __saturatef(input[index3 + 1]) * 255; // g
        ucharInput[index3 + 2] = __saturatef(input[index3 + 2]) * 255; // b
    }
}