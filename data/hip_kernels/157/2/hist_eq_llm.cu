#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define HISTOGRAM_LENGTH 256

__global__ void hist_eq(unsigned char * deviceCharImg, float * output, float* cdf, float cdfmin, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize thread index calculation

    if(i < size)
    {
        float val = deviceCharImg[i]; // Cache repeated memory access
        deviceCharImg[i] = min(max(255 * (cdf[val] - cdfmin) / (1 - cdfmin), 0.0f), 255.0f); // Simplify expression with cached value

        output[i] = deviceCharImg[i] / 255.0f; // Use float division directly
    }
}