#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define HISTOGRAM_LENGTH 256

__global__ void convertToGrayScale(unsigned char * ucharImg, unsigned char * grayImg, int width, int height)
{
    // Calculate unique thread index
    int index = blockIdx.y * blockDim.y * width + blockIdx.x * blockDim.x + threadIdx.y * width + threadIdx.x;

    if (index < width * height) // Ensure index is within the image range
    {
        grayImg[index] = (unsigned char) (0.21*ucharImg[index*3] + 0.71*ucharImg[index*3 + 1] + 0.07*ucharImg[index*3 + 2]);
    }
}