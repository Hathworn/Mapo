#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    // Calculate the row index
    int numRows = blockIdx.y * blockDim.y + threadIdx.y;
    // Calculate the column index
    int numColumns = blockIdx.x * blockDim.x + threadIdx.x;

    if (numRows < numCRows && numColumns < numCColumns) {
        float Cval = 0.0;
        
        // Optimize loop iteration conditions
        for (int k = 0; k < numAColumns; ++k) {
            Cval += A[numRows * numAColumns + k] * B[k * numBColumns + numColumns];
        }

        C[numRows * numCColumns + numColumns] = Cval;
    }
}