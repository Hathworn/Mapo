#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeHistogram64Kernel(float *d_Histogram, float *d_PartialHistograms, uint histogramCount) {
    __shared__ uint data[MERGE_THREADBLOCK_SIZE];

    uint sum = 0;

    // Accumulate partial histograms by each thread
    for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE) {
        sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM64_BIN_COUNT];
    }

    data[threadIdx.x] = sum;
    __syncthreads(); // Ensure all threads have written to shared memory

    // Perform parallel reduction in shared memory
    for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
        __syncthreads(); // Synchronize to ensure all reads in this stride are complete
    }

    // Write the result for this block to global memory
    if (threadIdx.x == 0) {
        d_Histogram[blockIdx.x] = data[0];
    }
}