#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TAG_MASK 0xFFFFFFFFU

__global__ void mergeHistogram256Kernel(uint *d_Histogram, uint *d_PartialHistograms, uint histogramCount)
{
    // Calculate global thread index for processing
    uint globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;
    uint sum = 0;

    // Accumulate histogram values for this thread
    for (uint i = globalThreadIdx; i < histogramCount * HISTOGRAM256_BIN_COUNT; i += gridDim.x * blockDim.x)
    {
        sum += d_PartialHistograms[i];
    }

    // Use shared memory for reduction within the block
    extern __shared__ uint data[];
    data[threadIdx.x] = sum;
    __syncthreads();

    // Efficient parallel reduction
    for (uint stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride)
        {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0)
    {
        d_Histogram[blockIdx.x] = data[0];
    }
}