#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeHistogram256Kernel(uint *d_Histogram, uint *d_PartialHistograms, uint histogramCount)
{
    // Initialize sum with 0
    uint sum = 0;

    // Accumulate partial histograms
    for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)
    {
        sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM256_BIN_COUNT];
    }

    // Allocate shared memory
    __shared__ uint data[MERGE_THREADBLOCK_SIZE];
    data[threadIdx.x] = sum;

    // Perform parallel reduction
    __syncthreads(); // Reduce synchronization overhead by having a single barrier
    for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride)
        {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
        __syncthreads(); // Ensure all updates to shared data are visible
    }

    // Write the result of this thread block to d_Histogram
    if (threadIdx.x == 0)
    {
        d_Histogram[blockIdx.x] = data[0];
    }
}