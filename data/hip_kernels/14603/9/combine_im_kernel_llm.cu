#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void combine_im_kernel(const float *A, const float *B, float *C, int numElements)
{
    // Calculate index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Constant computation outside of if-condition
    constexpr float B1 = 16.0f;
    constexpr float B2 = 16.0f;
    constexpr float denominator = B1 * B2 - 1;

    // Process only when index i is within bounds
    if (i < numElements)
    {
        // Optimize by removing redundant multiplication
        float temp = A[i] + B[i] * (B2 - 1);
        C[i] = B1 * temp / denominator;
    }
}