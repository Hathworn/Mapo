#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeHistogram64Kernel(uint *d_Histogram, uint *d_PartialHistograms, uint histogramCount)
{
    __shared__ uint data[MERGE_THREADBLOCK_SIZE];

    uint sum = 0;

    // Unrolling loop for performance improvement
    for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE * 2)
    {
        sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM64_BIN_COUNT];
        if (i + MERGE_THREADBLOCK_SIZE < histogramCount)
        {
            sum += d_PartialHistograms[blockIdx.x + (i + MERGE_THREADBLOCK_SIZE) * HISTOGRAM64_BIN_COUNT];
        }
    }

    data[threadIdx.x] = sum;

    // Optimized parallel reduction
    for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (threadIdx.x < stride)
        {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
    }

    if (threadIdx.x == 0)
    {
        d_Histogram[blockIdx.x] = data[0];
    }
}