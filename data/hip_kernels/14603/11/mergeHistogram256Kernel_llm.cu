#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeHistogram256Kernel(float *d_Histogram, float *d_PartialHistograms, uint histogramCount) 
{
    // Calculate partial sum using shared memory reduction
    __shared__ uint data[MERGE_THREADBLOCK_SIZE];
    
    uint sum = 0;
    for (uint i = threadIdx.x; i < histogramCount; i += MERGE_THREADBLOCK_SIZE)
    {
        sum += d_PartialHistograms[blockIdx.x + i * HISTOGRAM256_BIN_COUNT];
    }

    // Store sum in shared memory
    data[threadIdx.x] = sum;

    // Efficient reduction using shared memory
    __syncthreads();
    for (uint stride = MERGE_THREADBLOCK_SIZE / 2; stride > 0; stride >>= 1) 
    {
        if (threadIdx.x < stride) 
        {
            data[threadIdx.x] += data[threadIdx.x + stride];
        }
        __syncthreads(); // Wait for all threads before next stride
    }

    // Write result to global memory
    if (threadIdx.x == 0) 
    {
        d_Histogram[blockIdx.x] = data[0];
    }
}