#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_vdiv_kernel(int n, float *a, float *b, float *y) {
    // Compute the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride to allow each thread to process multiple elements if necessary
    int stride = blockDim.x * gridDim.x;
    
    for (; i < n; i += stride) {
        y[i] = a[i] / b[i];  // Element-wise division
    }
}