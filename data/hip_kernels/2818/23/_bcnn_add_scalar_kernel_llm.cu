#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_add_scalar_kernel(int n, float a, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    int grid_size = blockDim.x * gridDim.x; // Calculate the grid size

    for (; i < n; i += grid_size) { // Loop over elements to maximize memory access
        y[i] += a;
    }
}