#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _mean_variance_backward_kernel(float *x, float *grad, float *mean, float *var, int b, int c, int wxh, float *mean_diff, float *var_diff) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified global index calculation

    if (i >= c) return;

    // Register allocation to reduce global memory access
    float mean_diff_local = 0.0f;
    float var_diff_local = 0.0f;

    for (int j = 0; j < b; ++j) {
        for (int k = 0; k < wxh; ++k) {
            int ind = j * c * wxh + i * wxh + k;
            mean_diff_local += grad[ind];
            var_diff_local += grad[ind] * (x[ind] - mean[i]);
        }
    }

    // Use local variables to reduce redundant calculations and improve precision
    float inv_sqrt_var = rsqrtf(var[i] + 1e-5f);
    mean_diff[i] = mean_diff_local * (-inv_sqrt_var);
    var_diff[i] = var_diff_local * -0.5f / (var[i] * var[i] * inv_sqrt_var + 1e-5f);
}