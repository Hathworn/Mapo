#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bcnn_cuda_axpy_strided_kernel(int n, int num_batches, float a, float *x, float *y, int dst_stride, int src_stride, int x_c, int x_h, int x_w, int y_c, int y_h, int y_w, int min_c, int min_h, int min_w) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified calculation of thread id
    if (id >= n) return;

    int i, j, k, b;
    int remainder = id;

    // Unroll divisions to improve performance
    b = remainder / (min_c * min_h * min_w);
    remainder -= b * (min_c * min_h * min_w);
    k = remainder / (min_h * min_w);
    remainder -= k * (min_h * min_w);
    j = remainder / min_w;
    i = remainder % min_w;

    // Precompute common expressions
    int b_offset_y = y_c * b;
    int b_offset_x = x_c * b;

    int dst_int = i * dst_stride + y_w * (j * dst_stride + y_h * (b_offset_y + k));
    int src_ind = i * src_stride + x_w * (j * src_stride + x_h * (b_offset_x + k));

    y[dst_int] += a * x[src_ind];
}