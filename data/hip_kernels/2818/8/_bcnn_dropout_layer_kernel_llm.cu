#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_dropout_layer_kernel(float *input, int size, float *rand, float prob, float scale) {
    // Optimize to use shared memory and warp parallelism
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        input[id] *= (rand[id] >= prob) * scale;
    }
}