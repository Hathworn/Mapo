#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_backward_depthwise_conv_data_kernel(int nthreads, float *dst_grad, float *weight_data, int batch_size, const int channels, int dst_h, int dst_w, const int src_h, const int src_w, int kernel_sz, int stride, int pad, float *src_grad) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nthreads) {
        int n = i / channels / src_h / src_w;
        int c = (i / src_h / src_w) % channels;
        int h = (i / src_w) % src_h;
        int w = i % src_w;

        float *weight = weight_data + c * kernel_sz * kernel_sz;
        float value = 0.0f;

        for (int kh = 0; kh < kernel_sz; ++kh) {
            for (int kw = 0; kw < kernel_sz; ++kw) {
                int h_out_s = h + pad - kh;
                int w_out_s = w + pad - kw;
                
                // Ensure both stride conditions met
                if (h_out_s % stride == 0 && w_out_s % stride == 0) {
                    int h_out = h_out_s / stride;
                    int w_out = w_out_s / stride;
                    
                    // Check bounds for dst_grad access
                    if (h_out >= 0 && h_out < dst_h && w_out >= 0 && w_out < dst_w) {
                        int offset = ((n * channels + c) * dst_h + h_out) * dst_w + w_out;
                        value += weight[kh * kernel_sz + kw] * dst_grad[offset];
                    }
                }
            }
        }
        
        // Update src_grad at current index
        src_grad[i] += value;
    }
}