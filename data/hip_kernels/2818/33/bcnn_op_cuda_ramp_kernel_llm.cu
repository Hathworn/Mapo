#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bcnn_op_cuda_ramp_kernel(int n, float *x, float *y) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Stride for looping over array elements
    int stride = blockDim.x * gridDim.x;
    // Loop over elements to handle larger data sizes
    for (; i < n; i += stride) {
        y[i] = x[i] * (x[i] > 0) + 0.1 * x[i];
    }
}