#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_op_cuda_relu_kernel(int n, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    if (i < n) {
        y[i] = fmaxf(x[i], 0.0f);  // Use fmaxf for better performance
    }
}