#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using 1D grid and block
__global__ void _bcnn_vadd_kernel(int n, float *a, float *b, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = a[i] + b[i];
    }
}