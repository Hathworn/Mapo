#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _norm_backward_kernel(float *x, float *mean, float *var, float *mean_diff, float *var_diff, int b, int c, int wxh, float *grad) {
    int ind = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation
    int stride = blockDim.x * gridDim.x; // Calculate stride for loop
    int total = b * c * wxh;
  
    for (; ind < total; ind += stride) { // Use loop for processing elements
        int j = (ind / wxh) % c;
        float inv_sqrt_var = 1.0f / sqrtf(var[j] + 0.00001f); // Avoid recalculating
        grad[ind] = grad[ind] * inv_sqrt_var +
                    var_diff[j] * 2.0f * (x[ind] - mean[j]) / (wxh * b) +
                    mean_diff[j] / (wxh * b);
    }
}