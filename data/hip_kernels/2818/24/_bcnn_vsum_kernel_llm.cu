#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_vsum_kernel(int n, float *x, float *sum) {
    // Use shared memory to reduce global memory operations
    __shared__ float local_sum;

    // Initialize shared memory
    if (threadIdx.x == 0) local_sum = 0.0f;
    __syncthreads();

    // Calculate partial sum
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) atomicAdd(&local_sum, x[i]);

    // Accumulate the results from all threads to global sum
    __syncthreads();
    if (threadIdx.x == 0) atomicAdd(sum, local_sum);
}