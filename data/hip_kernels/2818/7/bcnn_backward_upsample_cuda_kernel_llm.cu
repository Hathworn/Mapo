#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bcnn_backward_upsample_cuda_kernel(size_t dst_sz, float *src, int w, int h, int c, int n, int size, float *dst) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;
    
    // Loop to handle more data if the kernel size is larger than the grid
    while (i < dst_sz) {
        int dst_idx = i;
        int dst_w = i % (w * size);
        i /= (w * size);
        int dst_h = i % (h * size);
        i /= (h * size);
        int dst_c = i % c;
        i /= c;
        int b = i % n;
        
        int in_w = dst_w / size;
        int in_h = dst_h / size;
        int src_idx = b * w * h * c + dst_c * w * h + in_h * w + in_w;
        
        atomicAdd(&src[src_idx], dst[dst_idx]); // Use atomicAdd for safe parallel accumulation
        
        i += stride; // Move to next index based on stride
    }
}