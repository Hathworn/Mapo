#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bcnn_grad_scales_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates) {
    __shared__ float part[BCNN_CUDA_THREADS];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f;

    // Unrolling the inner loop to maximize throughput and minimize overhead
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BCNN_CUDA_THREADS) {
            int index = i + size * (filter + n * b);
            sum += delta[index] * x_norm[index];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Using parallel reduction for optimized sum accumulation
    if (p < 128) part[p] += part[p + 128];
    __syncthreads();
    if (p < 64) part[p] += part[p + 64];
    __syncthreads();
    if (p < 32) {
        part[p] += part[p + 32];
        part[p] += part[p + 16];
        part[p] += part[p + 8];
        part[p] += part[p + 4];
        part[p] += part[p + 2];
        part[p] += part[p + 1];
    }

    // The first thread in each block sums the result
    if (p == 0) {
        scale_updates[filter] += part[0];
    }
}