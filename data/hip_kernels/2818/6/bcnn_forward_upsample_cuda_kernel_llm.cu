#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bcnn_forward_upsample_cuda_kernel(size_t dst_sz, float *src, int w, int h, int c, int n, int size, float *dst) {
    size_t i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= dst_sz) {
        return;
    }

    // Precompute common terms to reduce repetitive calculations
    int wh_size = w * size;
    int hh_size = h * size;
    int wh = w * h;
    int ch = c * wh;

    // Compute indices
    int b = i / (ch * size * size);
    int remainder = i % (ch * size * size);
    int dst_c = remainder / (wh_size * hh_size);
    remainder %= (wh_size * hh_size);
    int dst_h = remainder / wh_size;
    int dst_w = remainder % wh_size;

    // Calculate source indices
    int src_w = dst_w / size;
    int src_h = dst_h / size;
    int src_idx = b * ch + dst_c * wh + src_h * w + src_w;

    // Update destination
    dst[i] += src[src_idx];
}