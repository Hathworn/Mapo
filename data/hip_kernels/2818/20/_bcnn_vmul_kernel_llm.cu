#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_vmul_kernel(int n, float *a, float *b, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index efficiently
    int stride = blockDim.x * gridDim.x; // Calculate total stride

    // Utilize the stride for coalesced memory access
    for (; i < n; i += stride) {
        y[i] = a[i] * b[i]; // Perform vector multiplication
    }
}