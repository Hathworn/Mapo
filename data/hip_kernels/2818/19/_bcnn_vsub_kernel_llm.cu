#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_vsub_kernel(int n, float *a, float *b, float *y) {
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride to allow for flexible grid sizes
    int stride = blockDim.x * gridDim.x;
    // Perform vector subtraction with stride
    for (int index = i; index < n; index += stride) {
        y[index] = a[index] - b[index];
    }
}