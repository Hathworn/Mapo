#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_forward_softmax_layer_kernel(int n, int batch, float *input, float *output) {
    // Using shared memory for intermediate computations
    extern __shared__ float shared_mem[];

    int tid = threadIdx.x;
    int b = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + tid;

    if (b >= batch) {
        return;
    }

    // Calculate maxf using a single loop
    float maxf = -INFINITY;
    for (int i = tid; i < n; i += blockDim.x) {
        float val = input[i + b * n];
        if (val > maxf) {
            maxf = val;
        }
    }

    // Reduce maxf across the block
    shared_mem[tid] = maxf;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_mem[tid] = fmaxf(shared_mem[tid], shared_mem[tid + s]);
        }
        __syncthreads();
    }
    maxf = shared_mem[0];

    // Calculate sum exponential
    float sum = 0.f;
    for (int i = tid; i < n; i += blockDim.x) {
        sum += expf(input[i + b * n] - maxf);
    }

    // Reduce sum
    shared_mem[tid] = sum;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_mem[tid] += shared_mem[tid + s];
        }
        __syncthreads();
    }
    sum = shared_mem[0];

    sum = (sum != 0) ? maxf + logf(sum) : maxf - 100.f;

    // Write output
    for (int i = tid; i < n; i += blockDim.x) {
        output[i + b * n] = expf(input[i + b * n] - sum);
    }
}