#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_pow_kernel(int n, float *x, float a, float *y) {
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterate with stride for all elements
    for (; i < n; i += blockDim.x * gridDim.x) {
        y[i] = powf(x[i], a);  // Use powf for single precision
    }
}