#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_op_cuda_tanh_kernel(int n, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation using 1D grid
    int gridStride = gridDim.x * blockDim.x; // Compute stride for grid-wide loop
    for (; i < n; i += gridStride) { // Use grid stride loop for better parallelization
        float exp_val = expf(2 * x[i]); // Compute exp once to optimize performance
        y[i] = (exp_val - 1) / (exp_val + 1);
    }
}