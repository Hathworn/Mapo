#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta) {
    const int threads = 512;
    __shared__ float local[threads];

    // Calculate global index for current thread
    int global_id = blockIdx.x * spatial + threadIdx.x;
    local[threadIdx.x] = 0;

    // Use single loop with stride pattern for coalesced memory access
    for (int j = 0; j < batch; ++j) {
        for (int i = global_id; i < batch * spatial * filters; i += blockDim.x * gridDim.x) {
            local[threadIdx.x] += (i < batch * spatial * filters) ? delta[i] : 0;
        }
    }

    __syncthreads();

    // Reduction using a single thread
    if (threadIdx.x == 0) {
        float sum = 0;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        mean_delta[blockIdx.x] = sum * (-1.f / sqrtf(variance[blockIdx.x] + .00001f));
    }
}