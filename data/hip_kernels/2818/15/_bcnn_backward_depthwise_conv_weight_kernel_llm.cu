#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_backward_depthwise_conv_weight_kernel(int nthreads, float *dst_grad, float *src_data, int batch_size, const int channels, int dst_h, int dst_w, const int src_h, const int src_w, int kernel_sz, int stride, int pad, float *weight_diff) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nthreads) return;  // Early exit for out-of-bound threads

    int n = i / (channels * src_h * src_w);
    int c = (i / (src_h * src_w)) % channels;
    int h = (i / src_w) % src_h;
    int w = i % src_w;
    
    float *p_weight_diff = weight_diff + c * kernel_sz * kernel_sz;

    for (int kh = 0; kh < kernel_sz; ++kh) {
        for (int kw = 0; kw < kernel_sz; ++kw) {
            int h_out_s = h + pad - kh;
            int w_out_s = w + pad - kw;
            
            // Ensure h_out_s and w_out_s are divisible by stride
            if ((h_out_s % stride) == 0 && (w_out_s % stride) == 0) {
                int h_out = h_out_s / stride;
                int w_out = w_out_s / stride;
                
                // Check bounds
                if (h_out >= 0 && h_out < dst_h && w_out >= 0 && w_out < dst_w) {
                    int offset = ((n * channels + c) * dst_h + h_out) * dst_w + w_out;
                    // Accumulate weight gradient
                    atomicAdd(p_weight_diff, src_data[i] * dst_grad[offset]);
                }
            }
            ++p_weight_diff;
        }
    }
}