#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _norm_forward_kernel(float *x, float *mean, float *variance, int b, int c, int wxh) {
    int ind = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified calculation of 'ind'
    int num_elements = b * c * wxh;  // Pre-calculation for loop boundary

    for (int idx = ind; idx < num_elements; idx += blockDim.x * gridDim.x) {  // Loop over elements with grid-stride
        int j = (idx / wxh) % c;  // Calculate 'j' for current element
        x[idx] = (x[idx] - mean[j]) / (sqrt(variance[j] + 0.000001f));  // Perform normalization
    }
}