#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean) {
    const int threads = 512;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    int stride = spatial * filters;

    // Initialize shared memory
    local[id] = 0;

    // Accumulate sum in shared memory
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * stride + filter * spatial + i;
            local[id] += x[index];
        }
    }

    __syncthreads();

    // Reduce sum in shared memory
    if (id < 256) local[id] += local[id + 256];
    __syncthreads();
    if (id < 128) local[id] += local[id + 128];
    __syncthreads();
    if (id < 64) local[id] += local[id + 64];
    __syncthreads();

    if (id < 32) {
        local[id] += local[id + 32];
        local[id] += local[id + 16];
        local[id] += local[id + 8];
        local[id] += local[id + 4];
        local[id] += local[id + 2];
        local[id] += local[id + 1];
    }

    if (id == 0) {
        mean[filter] = local[0] / (spatial * batch);  // Compute mean
    }
}