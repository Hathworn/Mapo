#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta) {
    const int threads = 512;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Use a single loop iteration and reduce shared memory access
    for (int j = 0; j < batch * spatial; j += threads) {
        int index = j * filters + filter * spatial + id;

        if (j + id < batch * spatial) {
            local[id] += delta[index] * (x[index] - mean[filter]);
        }
    }

    __syncthreads();

    // Perform reduction using parallel threads
    if (id < threads / 2) {
        local[id] += local[id + threads / 2];
    }
    __syncthreads();

    // Calculate final variance delta in a single thread
    if (id == 0) {
        float sum = 0;
        for (int i = 0; i < threads / 2; ++i) {
            sum += local[i];
        }
        variance_delta[filter] = -.5f * sum * powf(variance[filter] + .00001f, -1.5f);
    }
}