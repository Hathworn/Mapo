#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _mean_variance_forward_kernel(float *x, int b, int c, int wxh, float *mean, float *var) {
    float scale = 1.0f / (b * wxh);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= c) return;
    
    float temp_mean = 0.0f;
    float temp_var = 0.0f;
    
    // Calculate the sum and sum of squares in parallel
    for (int j = 0; j < b; ++j) {
        for (int k = 0; k < wxh; ++k) {
            int ind = j * c * wxh + i * wxh + k;
            temp_mean += x[ind];
            temp_var += x[ind] * x[ind];
        }
    }
    
    // Compute mean and variance
    mean[i] = temp_mean * scale;
    var[i] = temp_var * scale - mean[i] * mean[i];
}