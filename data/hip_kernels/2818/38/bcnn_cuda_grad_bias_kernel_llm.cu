#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bcnn_cuda_grad_bias_kernel(float *grad_bias, float *grad_data, int num_channels, int spatial_size) {
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int channel = blockIdx.y;
    int batch_size = blockIdx.z;

    // Use shared memory to accumulate the gradient across spatial dimensions
    extern __shared__ float shared_grad[];
    float local_grad = 0.0f;

    if (offset < spatial_size) {
        local_grad = grad_data[(batch_size * num_channels + channel) * spatial_size + offset];
    }

    // Reduce within block
    shared_grad[threadIdx.x] = local_grad;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shared_grad[threadIdx.x] += shared_grad[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Accumulate the results into global memory by a single thread in the block
    if (threadIdx.x == 0) {
        atomicAdd(&grad_bias[channel], shared_grad[0]);
    }
}