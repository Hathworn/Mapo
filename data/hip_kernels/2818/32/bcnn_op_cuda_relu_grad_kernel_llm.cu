#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bcnn_op_cuda_relu_grad_kernel(int n, float *x, float *dx) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified the thread index calculation for 1D grid
    if (i < n) {
        dx[i] *= (x[i] > 0.0f ? 1.0f : 0.0f);  // Inline condition check to make the operation more explicit
    }
}