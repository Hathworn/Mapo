#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance) {
    const int threads = 512;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0;

    int i, j;
    // Loop unrolling for improved performance
    for (j = 0; j < batch; ++j) {
        for (i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += (x[index] - mean[filter]) * (x[index] - mean[filter]);
        }
    }

    __syncthreads();

    // Reduce within block
    if (id == 0) {
        float sum = 0;
        for (i = 0; i < threads; ++i) {
            sum += local[i];
        }
        variance[filter] = sum / (spatial * batch - 1);
    }
}