#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_forward_depthwise_conv_weight_kernel(int nthreads, float *src_data, float *weight_data, int channels, int dst_h, int dst_w, int src_h, int src_w, int kernel_sz, int stride, int pad, float *dst_data) {
    // Get the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nthreads) return; // Boundary check

    int n = i / (channels * dst_h * dst_w);
    int c = (i / (dst_h * dst_w)) % channels;
    int h = (i / dst_w) % dst_h;
    int w = i % dst_w;
    float *weight = weight_data + c * kernel_sz * kernel_sz;
    float value = 0.0f;
    
    // Loop over kernel size
    for (int kh = 0; kh < kernel_sz; ++kh) {
        int h_in = -pad + h * stride + kh;
        if (h_in >= 0 && h_in < src_h) { // Check vertical bounds
            for (int kw = 0; kw < kernel_sz; ++kw) {
                int w_in = -pad + w * stride + kw;
                if (w_in >= 0 && w_in < src_w) { // Check horizontal bounds
                    int offset = ((n * channels + c) * src_h + h_in) * src_w + w_in;
                    value += weight[kh * kernel_sz + kw] * src_data[offset];
                }
            }
        }
    }

    dst_data[i] = value;
}