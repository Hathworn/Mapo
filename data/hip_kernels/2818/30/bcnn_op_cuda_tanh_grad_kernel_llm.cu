#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_op_cuda_tanh_grad_kernel(int n, float *x, float *dx) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify index calculation
    if (i < n) {
        float val = x[i];  // Cache x[i] to reduce memory access
        dx[i] *= (1 - val * val);  // Utilize cached value
    }
}