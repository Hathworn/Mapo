#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Compute global index for each thread
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Memory bounds checking for safety
    if (i < blockDim.x * gridDim.x) {
        c[i] = a[i] + b[i];
    }
}