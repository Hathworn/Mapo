#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TestDimensions()
{
    int const num_threads_per_block = blockDim.x * blockDim.y * blockDim.z;
    int const thread_id = blockIdx.x * num_threads_per_block + 
                          threadIdx.z * blockDim.x * blockDim.y +
                          threadIdx.y * blockDim.x + 
                          threadIdx.x;  // Simplified thread_id calculation

    if(thread_id == 0)
    {
        printf("gridDim   = x: %6d / y: %6d / z: %6d\r\n", gridDim.x, gridDim.y, gridDim.z);
        printf("blockDim  = x: %6d / y: %6d / z: %6d\r\n", blockDim.x, blockDim.y, blockDim.z);
        printf("num_threads_per_block: %6d\r\n", num_threads_per_block);
        int const num_blocks = gridDim.x * gridDim.y * gridDim.z;
        int const threads_stride = num_threads_per_block * num_blocks;  // Calculate only in thread 0
        printf("num_blocks           : %6d\r\n", num_blocks);
        printf("threads_stride       : %6d\r\n", threads_stride);
    }
    
    printf("tidx | %6d | %6d | %6d | bidx | %6d | %6d | %6d | "
           "gdim | %6d | %6d | %6d | bdim | %6d | %6d | %6d | "
           "thread_id | %6d |\r\n",
           threadIdx.x, threadIdx.y, threadIdx.z,
           blockIdx.x, blockIdx.y, blockIdx.z,
           gridDim.x, gridDim.y, gridDim.z,
           blockDim.x, blockDim.y, blockDim.z, thread_id);
}