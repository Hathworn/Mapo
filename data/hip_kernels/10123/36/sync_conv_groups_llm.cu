#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() {
    // Use a more efficient synchronization method:
    __syncthreads();  // Synchronize threads within each block efficiently
}