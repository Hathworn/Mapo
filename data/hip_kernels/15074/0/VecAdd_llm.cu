#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAdd(double* A, double* B, double* C, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use block and thread index for global indexing
    if (i < n) // Check bounds to avoid accessing out of range
    {
        C[i] = A[i] + B[i];
    }
}