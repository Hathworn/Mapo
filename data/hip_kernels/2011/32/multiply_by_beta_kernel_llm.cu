#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiply_by_beta_kernel(float *input, float *output, float beta)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Use flat indexing to improve memory access efficiency
    long int id = ((k * c_Size.y + j) * c_Size.x + i);

    if (id < c_Size.x * c_Size.y * c_Size.z)  // Check within bounds for a flat array
        output[id] = input[id] * beta;
}