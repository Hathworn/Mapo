#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void magnitude_threshold_kernel(float * grad_x, float * grad_y, float * grad_z, float gamma)
{
    // Calculate linear thread index
    unsigned long long int id = (blockIdx.z * blockDim.z + threadIdx.z) * (c_Size.y * c_Size.x) 
                                + (blockIdx.y * blockDim.y + threadIdx.y) * c_Size.x 
                                + (blockIdx.x * blockDim.x + threadIdx.x);

    // Terminate if index is out of bounds
    if (blockIdx.x * blockDim.x + threadIdx.x >= c_Size.x || 
        blockIdx.y * blockDim.y + threadIdx.y >= c_Size.y || 
        blockIdx.z * blockDim.z + threadIdx.z >= c_Size.z)
        return;

    // Compute norm and apply threshold logic
    float norm = sqrt(grad_x[id] * grad_x[id] + grad_y[id] * grad_y[id] + grad_z[id] * grad_z[id]);
    if (norm > gamma)
    {
        float ratio = gamma / norm;
        grad_x[id] *= ratio;
        grad_y[id] *= ratio;
        grad_z[id] *= ratio;
    }
}