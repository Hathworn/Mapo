#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gradient_and_subtract_kernel(float *in, float *grad_x, float *grad_y, float *grad_z)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Combine boundary checks for better readability
    if (i < c_Size.x && j < c_Size.y && k < c_Size.z)
    {
        long int id = (k * c_Size.y + j) * c_Size.x + i;

        // Simplify index calculations
        if (i != (c_Size.x - 1))
        {
            long int id_x = id + 1;
            grad_x[id] -= (in[id_x] - in[id]) / c_Spacing.x;
        }
        if (j != (c_Size.y - 1))
        {
            long int id_y = id + c_Size.x;
            grad_y[id] -= (in[id_y] - in[id]) / c_Spacing.y;
        }
        if (k != (c_Size.z - 1))
        {
            long int id_z = id + c_Size.y * c_Size.x;
            grad_z[id] -= (in[id_z] - in[id]) / c_Spacing.z;
        }
    }
}