#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_volume_to_constant(float * out, float value)
{
    // Combine i, j, k calculation to improve efficiency
    unsigned long id = (blockIdx.z * blockDim.z + threadIdx.z) * c_Size.y * c_Size.x +
                       (blockIdx.y * blockDim.y + threadIdx.y) * c_Size.x +
                       (blockIdx.x * blockDim.x + threadIdx.x);

    // Use early return to reduce thread work
    if (blockIdx.x * blockDim.x + threadIdx.x >= c_Size.x || 
        blockIdx.y * blockDim.y + threadIdx.y >= c_Size.y || 
        blockIdx.z * blockDim.z + threadIdx.z >= c_Size.z)
        return;

    out[id] = value;
}