#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BOOL int
#define TRUE 1
#define FALSE 0
#define populationSize 128
#define chromosomeSize 10
#define maxGeneration 500
#define crossRate 0.8
#define mutationRate 0.01
#define eliteCount 0.05*populationSize

float LB[10] = {0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5}; //lower bound
float UB[10] = {5, 4, 5, 4, 5, 5, 5, 5, 5, 4}; //upper bound
float *a;  //Tzaihe
float *aa;  //yingliK
float *aaa; //Tyingli
int aRow;
int aaaRow;
float Dysum[9];

__device__ float c_LB[10]; //lower bound
__device__ float c_UB[10]; //upper bound
__device__ float *c_a;  //Tzaihe
__device__ float *c_aa;  //yingliK
__device__ float *c_aaa; //Tyingli
__device__ int c_aRow;
__device__ int c_aaaRow;
__device__ float c_Dysum[9];

float bestFitnessOfGen; //每一代的最优适应度
int bestIndexOfGen; //每一代的最优适应度位置
float aveFitnessOfGen[maxGeneration]; //每一代的平均最优适应度

float fval; //最终最优适应度
int G; //取得最终最优适应度的迭代次数

__global__ void mutationFcn(float *populationArray, hiprandState_t *states) {
    int idx = threadIdx.x;
    hiprandState_t s = states[idx];
    float ss = hiprand_uniform(&s);
    
    float scale = 0.5 * (1 - 0.75 * idx / maxGeneration); //Calculate scale more efficiently

    // Check if the current individual should mutate
    if (ss < mutationRate) {
        for (int j = 0; j < chromosomeSize; j++) {
            hiprandState_t t = states[idx];
            int tt = hiprand(&t);

            // Check if the current chromosome should mutate
            if (tt % 2 != 0) {
                float tmpChromosome;
                do {
                    tmpChromosome = populationArray[idx * chromosomeSize + j] + scale * (c_UB[j] - c_LB[j]) * hiprand_uniform(&s);
                    // Check if out of bounds
                } while (tmpChromosome > c_UB[j] || tmpChromosome < c_LB[j]);
                populationArray[idx * chromosomeSize + j] = tmpChromosome;
            }
        }
    }
}