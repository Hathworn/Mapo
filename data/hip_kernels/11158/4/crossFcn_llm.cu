#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BOOL int
#define TRUE 1
#define FALSE 0
#define populationSize 128
#define chromosomeSize 10
#define maxGeneration 500
#define crossRate 0.8f
#define mutationRate 0.01f
#define eliteCount (0.05 * populationSize)

float LB[10] = {0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f, 0.5f}; //lower bound
float UB[10] = {5.0f, 4.0f, 5.0f, 4.0f, 5.0f, 5.0f, 5.0f, 5.0f, 5.0f, 4.0f}; //upper bound
float *a;  //Tzaihe
float *aa;  //yingliK
float *aaa; //Tyingli
int aRow;
int aaaRow;
float Dysum[9];

__device__ float c_LB[10]; //lower bound
__device__ float c_UB[10]; //upper bound
__device__ float *c_a;  //Tzaihe
__device__ float *c_aa;  //yingliK
__device__ float *c_aaa; //Tyingli
__device__ int c_aRow;
__device__ int c_aaaRow;
__device__ float c_Dysum[9];

float bestFitnessOfGen; //每一代的最优适应度
int bestIndexOfGen; //每一代的最优适应度位置
float aveFitnessOfGen[maxGeneration]; //每一代的平均最优适应度

float fval; //最终最优适应度
int G; //取得最终最优适应度的迭代次数

__global__ void crossFcn(float *populationArray, hiprandState_t *states) {
    int idx = threadIdx.x;
    hiprandState_t s = states[idx];
    float randVal = hiprand_uniform(&s); // Generate random value once

    if (randVal < crossRate) { // Check if crossover should occur
        int offset = idx * chromosomeSize;
        int offsetMate = (idx + populationSize / 2) * chromosomeSize;
        for (int j = 0; j < chromosomeSize; j++) {
            if ((hiprand(&s) & 1) != 0) { // Randomly decide if genes should be crossed
                // Swap genes
                float tmp = populationArray[offset + j];
                populationArray[offset + j] = populationArray[offsetMate + j];
                populationArray[offsetMate + j] = tmp;
            }
        }
    }
}