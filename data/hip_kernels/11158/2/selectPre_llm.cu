#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BOOL int
#define TRUE 1
#define FALSE 0
#define populationSize 128
#define chromosomeSize 10
#define maxGeneration 500
#define crossRate 0.8
#define mutationRate 0.01
#define eliteCount 0.05*populationSize

float LB[10] = {0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5, 0.5}; //lower bound
float UB[10] = {5, 4, 5, 4, 5, 5, 5, 5, 5, 4}; //upper bound
float *a;
float *aa;
float *aaa;
int aRow;
int aaaRow;
float Dysum[9];

__device__ float c_LB[10];
__device__ float c_UB[10];
__device__ float *c_a;
__device__ float *c_aa;
__device__ float *c_aaa;
__device__ int c_aRow;
__device__ int c_aaaRow;
__device__ float c_Dysum[9];

float bestFitnessOfGen;
int bestIndexOfGen;
float aveFitnessOfGen[maxGeneration];

float fval;
int G;

__global__ void selectPre(float *fitness, float *Fitness, float *tmpFitness, float *populationArray, float *tmpPopulationArray){
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Ensure unique global index
    if (idx < populationSize) { // Boundary check for the population size
        Fitness[idx] = 1.0f / fitness[idx]; // Calculate Fitness value
        tmpFitness[idx] = fitness[idx]; // Copy fitness to tmpFitness
        // Use memcpy for copying chromosome
        memcpy(&tmpPopulationArray[idx * chromosomeSize], &populationArray[idx * chromosomeSize], chromosomeSize * sizeof(float));
    }
}