#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose(size_t sz, float_t* src, float_t* dest, size_t src_width, size_t src_height)
{
    // Calculate index using block and thread indices
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if computed index is within bounds
    if(index < sz)
    {
        // Calculate row (i) and column (j) in source matrix
        size_t i = index / src_width;
        size_t j = index % src_width;

        // Calculate transposed index for destination matrix
        size_t dest_index = j * src_height + i;

        // Perform transposition by assigning source to destination
        dest[dest_index] = src[index];
    }
}