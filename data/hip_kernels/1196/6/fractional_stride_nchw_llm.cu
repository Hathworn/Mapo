#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fractional_stride_nchw(size_t num_values, size_t stride, float_t* src, float_t* dest, size_t ld_src, size_t ld_dest)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds before computation
    if (index < num_values)
    {
        // Use integer division and modulo to compute frame_id in one line
        size_t frame_id = (index / ld_src) * ld_dest + (stride * (index % ld_src));
        
        // Move data from source to destination for the calculated frame_id
        dest[frame_id] = src[index];
    }
}