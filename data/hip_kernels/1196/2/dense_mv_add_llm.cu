#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dense_mv_add(size_t sz, float_t* src, float_t* dest)
{
    // Calculate global thread index
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use strided loop for better memory access pattern
    for (size_t i = index; i < sz; i += blockDim.x * gridDim.x) 
    {
        dest[i] += src[i];
    }
}