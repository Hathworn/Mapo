#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dense_add(size_t sz, float_t* src, float_t* dest)
{
    size_t destIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(destIndex < sz)
    {
        // Use 'destIndex' directly since 'srcIndex' always equals 'threadIdx.x'
        dest[destIndex] += src[threadIdx.x];
    }
}