#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reshape(size_t num_values, float_t* src, float_t* dest, size_t ld_src, size_t ld_dest)
{
    // Using shared memory to reduce global memory access and increase cache hits
    __shared__ float_t sharedSrc[256]; // Adjust size according to your needs and hardware
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t localIndex = threadIdx.x;
    if (index < num_values)
    {
        size_t src_index = (index / ld_dest) * ld_src + index % ld_dest;
        sharedSrc[localIndex] = src[src_index]; // Load data to shared memory
        __syncthreads();  // Ensure all threads have written to shared memory before reading

        dest[index] = sharedSrc[localIndex]; // Copy from shared memory to global memory
    }
}