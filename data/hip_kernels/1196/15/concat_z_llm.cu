#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void concat_z(size_t sz, float_t* src, float_t* dest, float_t* z, size_t stride)
{
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;

    // Use one conditional statement to improve readability and reduce branching
    if (index < sz)
    {
        dest[index] = (index >= stride) ? src[index - stride] : z[index];
    }
}