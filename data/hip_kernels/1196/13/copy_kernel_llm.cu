#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(size_t sz, float_t* src, float_t* dest) 
{
    // Calculate global index
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    // Determine stride through grid of blocks
    size_t stride = blockDim.x * gridDim.x;

    // Use strided loop for optimal memory access
    for(size_t i = index; i < sz; i += stride) 
    {
        dest[i] = src[i];
    }
}