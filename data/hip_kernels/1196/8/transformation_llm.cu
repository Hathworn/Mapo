#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transformation(size_t num_values, float_t* src, float_t* dest, size_t ld_src, size_t ld_dest)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for coalesced access
    __shared__ float_t shared_src[256];  // Adjust size as necessary (change 256 if needed)

    if (index < num_values)
    {
        // Load data into shared memory
        shared_src[threadIdx.x] = src[index];
        __syncthreads();

        // Calculate destination index
        size_t dest_index = (index / ld_src) * ld_src + ((index % ld_src) % 8) * ld_dest + (index % ld_src) / 8;

        // Store data from shared memory to global memory
        dest[dest_index] = shared_src[threadIdx.x];
    }
}