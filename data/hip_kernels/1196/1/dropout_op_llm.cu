#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dropout_op(size_t sz, float_t* random_nums, float_t* data, float_t drop_rate, float_t scale)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop for better efficiency and use a branchless approach
    if (index < sz)
    {
        float_t rand_val = random_nums[index];
        float_t mask = float_t(rand_val > drop_rate);
        data[index] *= mask * scale; // Use mask to conditionally multiply data[index] with scale, replacing dropped values with zero.
    }
}