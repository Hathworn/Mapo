#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void skip_res_add(size_t sz, float_t* f5, float* f1, float_t* skip_out_sum, size_t stride)
{
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;

    // Use loop unrolling to maximize memory throughput
    #pragma unroll 4
    for (size_t i = index; i < sz; i += blockDim.x * gridDim.x)
    {
        skip_out_sum[i] += f5[i + stride];
        f1[i] += f5[i];
    }
}