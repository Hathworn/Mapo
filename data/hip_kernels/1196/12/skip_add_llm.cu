#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void skip_add(size_t sz, float_t* f1, float* skip_out_sum)
{
    // Use a single variable to compute the global index, for clarity.
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;

    // Check boundary and perform addition in one step for efficiency.
    if (index < sz) {
        skip_out_sum[index] = skip_out_sum[index] + f1[index];
    }
}