#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_kernel(size_t sz, float_t* src, float_t* dest, size_t ld_src, size_t ld_dest)
{
    // Calculate the linear index and convert to 2D grid indices
    size_t j = blockIdx.x * blockDim.x + threadIdx.x;
    size_t i = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check bounds to avoid invalid memory accesses
    if(i < ld_src && j < ld_dest && (j * ld_dest + i) < sz)
    {
        // Transpose the matrix element
        dest[j * ld_dest + i] = src[i * ld_src + j];
    }
}