#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dense_add_conv(size_t sz, float_t* src, float_t* dest, size_t bias_dim)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use index modulo bias_dim directly in array access, 
    // to simplify and improve memory access pattern
    if (index < sz)
    {
        dest[index] += src[index % bias_dim];
    }
}