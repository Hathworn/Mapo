#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void affine_transform(size_t sz, float_t* audio, float_t* end_out, size_t stride)
{
    // Calculate global thread index
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimize memory access pattern
    if(index + stride < sz + stride)
    {
        float_t out_index = end_out[index];
        audio[index + stride] = (audio[index + stride] - out_index) / expf(end_out[index + stride]);
    }
}