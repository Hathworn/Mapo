#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void suma(int a, int b, int *c) {
    // Use blockIdx.x to manage multiple blocks if needed; avoid atomic if writing to a single memory location per block
    if (threadIdx.x == 0) { // Ensure only one thread performs the addition
        *c = a + b;
    }
}