#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpy(double *y, double *a, double *x, int m, int n) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    // Check if block index is within bounds
    if (bid < m) {
        // Each thread computes its part of the dot product
        if (tid < n) {
            dots_s[tid] = a[bid * n + tid] * x[tid];
        } else {
            dots_s[tid] = 0.0;  // Initialize unused part of shared memory to zero
        }
        __syncthreads();

        // Use parallel reduction to sum the results
        for (int stride = n / 2; stride > 0; stride /= 2) {
            if (tid < stride) {
                dots_s[tid] += dots_s[tid + stride];
            }
            __syncthreads();
        }

        // Only the first thread writes the result
        if (tid == 0) {
            y[bid] = dots_s[0];
        }
    }
}