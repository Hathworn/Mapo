#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpymm(double *y, double *a, double *b, int m, int n, int p) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    // Check if block index is within range
    if (bid < m) {
        // Load and compute partial results into shared memory
        for (int c = 0; c < p; c++) {
            if (tid < n) {
                dots_s[tid*p + c] = a[bid*n + tid] * b[tid*p + c];
            }
        }

        __syncthreads();

        // First thread in block reduces partial results
        if (tid == 0) {
            for (int c = 0; c < p; c++) {
                double sum = 0.0;
                for (int i = 0; i < n; i++) {
                    sum += dots_s[i*p + c];
                }
                y[bid*p + c] = sum; // Store result in global memory
            }
        }
    }
}