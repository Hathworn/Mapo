#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulSh(float *Md, float *Nd, float *Pd, const int WIDTH) {
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // Calculate thread id
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;
    float Pvalue = 0; // Initialize accumulation result to zero

    for (int m = 0; m < WIDTH / TILE_WIDTH; m++) {
        // Load data into shared memory
        Mds[threadIdx.y][threadIdx.x] = Md[row * WIDTH + (m * TILE_WIDTH + threadIdx.x)];
        Nds[threadIdx.y][threadIdx.x] = Nd[(m * TILE_WIDTH + threadIdx.y) * WIDTH + col];
        __syncthreads(); // Synchronize threads to ensure complete data load

        // Perform matrix multiplication on the tile
        for (int k = 0; k < TILE_WIDTH; k++) {
            Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        }
        __syncthreads(); // Synchronize before next tile
    }

    // Write final result to global memory
    Pd[row * WIDTH + col] = Pvalue;
}