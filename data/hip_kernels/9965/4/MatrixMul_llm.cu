#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMul( float *Md , float *Nd , float *Pd , const int WIDTH )
{
    // calculate thread id
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

    // Ensure within matrix bounds
    if (col < WIDTH && row < WIDTH) {
        float Pvalue = 0.0; // Use register to accumulate results
        for (int k = 0; k < WIDTH; k++) {
            Pvalue += Md[row * WIDTH + k] * Nd[k * WIDTH + col];
        }
        Pd[row * WIDTH + col] = Pvalue; // Store result
    }
}
```
