#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cube(float * d_out, float * d_in) {
    // Using blockIdx and blockDim for accessing global index in grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f * f;
}