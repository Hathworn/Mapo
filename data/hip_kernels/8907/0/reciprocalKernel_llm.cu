#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
============================================================================
Name        : Teste.cu
Author      :
Version     :
Copyright   : Your copyright notice
Description : CUDA compute reciprocals
============================================================================
*/

static void CheckCudaErrorAux(const char *, unsigned, const char *, hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__, __LINE__, #value, value)

/**
* CUDA kernel that computes reciprocal values for a given vector
*/
__global__ void reciprocalKernel(float *data, unsigned vectorSize) {
    // Calculate global index
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bounds and compute reciprocal for valid element
    if (idx < vectorSize) {
        data[idx] = 1.0f / data[idx]; // Use '1.0f' for float precision
    }
}