#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CUDA_THREADS_PER_BLOCK 1024

__global__ void Max_Interleaved_Addressing_Shared(float* data, int data_size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ float sdata[MAX_CUDA_THREADS_PER_BLOCK];

    // Load data into shared memory
    if (idx < data_size) {
        sdata[threadIdx.x] = data[idx];
    } else {
        sdata[threadIdx.x] = -FLT_MAX; // Handle out-of-bound threads
    }
    __syncthreads();

    // Perform reduction within shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            float lhs = sdata[threadIdx.x];
            float rhs = sdata[threadIdx.x + stride];
            sdata[threadIdx.x] = lhs < rhs ? rhs : lhs;
        }
        __syncthreads();
    }

    // Write result back to global memory
    if (threadIdx.x == 0) {
        data[blockIdx.x] = sdata[0];
    }
}