#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CUDA_THREADS_PER_BLOCK 1024

__global__ void Max_Interleaved_Addressing_Global(float* data, int data_size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimized: Use shared memory for block-level reduction
    extern __shared__ float shared_data[];

    if (idx < data_size) {
        shared_data[threadIdx.x] = data[idx];
    } else {
        shared_data[threadIdx.x] = -INFINITY; // Ensure there's no effect on max
    }
    __syncthreads();

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int index = 2 * stride * threadIdx.x;
        if (index + stride < blockDim.x) {
            shared_data[index] = max(shared_data[index], shared_data[index + stride]);
        }
        __syncthreads();
    }

    // Write the block's result back to global memory
    if (threadIdx.x == 0) {
        data[blockIdx.x] = shared_data[0];
    }
}