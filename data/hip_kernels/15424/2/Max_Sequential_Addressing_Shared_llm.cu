#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_CUDA_THREADS_PER_BLOCK 1024

__global__ void Max_Sequential_Addressing_Shared(float* data, int data_size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ float sdata[MAX_CUDA_THREADS_PER_BLOCK];

    // Load data into shared memory and ensure within bounds
    if (idx < data_size) {
        sdata[threadIdx.x] = data[idx];
    } else {
        sdata[threadIdx.x] = -FLT_MAX; // Use negative max for unused threads
    }
    __syncthreads();

    // Perform reduction using shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            float lhs = sdata[threadIdx.x];
            float rhs = sdata[threadIdx.x + stride];
            sdata[threadIdx.x] = lhs < rhs ? rhs : lhs; // Store max
        }
        __syncthreads();
    }

    // Write the result back to the first position of data
    if (threadIdx.x == 0) {
        data[blockIdx.x] = sdata[0]; // Store final block max
    }
}