#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global index for better scalability
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}