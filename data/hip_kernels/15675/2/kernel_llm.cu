#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use __device__ function to minimize kernel overhead
__device__ void printHello() {
    printf("hello world");
}

__global__ void kernel() {
    // Launch kernel for each thread block
    if (threadIdx.x == 0 && blockIdx.x == 0) { // Reduce unnecessary prints
        printHello();
    }
}