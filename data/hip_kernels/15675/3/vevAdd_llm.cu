#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vevAdd(int N, float *a, float *b, float *c)
{
    // Use shared memory for improved performance
    extern __shared__ float shared_a[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (idx < N)
    {
        shared_a[threadIdx.x] = a[idx];
        c[idx] = shared_a[threadIdx.x] + b[idx];
    }
}