#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil(int *in, int *out)
{
    __shared__ int sharedMem[BLOCK_SIZE + 2 * RADIUS];  // Allocate shared memory

    int localIdx = threadIdx.x + RADIUS;
    int globIdx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data into shared memory
    sharedMem[localIdx] = in[globIdx];
    
    // Load halo data
    if (threadIdx.x < RADIUS) {
        sharedMem[localIdx - RADIUS] = in[globIdx - RADIUS];
        sharedMem[localIdx + BLOCK_SIZE] = in[globIdx + BLOCK_SIZE];
    }
    
    __syncthreads();  // Synchronize threads to ensure all data is loaded

    int value = 0;
    for(int offset = -RADIUS; offset <= RADIUS; offset++)
        value += sharedMem[localIdx + offset];  // Use shared memory for computation
    
    out[globIdx] = value;
}