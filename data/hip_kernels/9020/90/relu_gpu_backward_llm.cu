#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use read-only cache for input data
    if (tid < N) {
        float input_val = indata[tid];
        float output_val = outgrad[tid];
        ingrad[tid] = input_val > 0 ? output_val : 0;
    }
}