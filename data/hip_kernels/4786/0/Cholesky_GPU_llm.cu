#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

const int MAX = 100;

__global__ void Cholesky_GPU(double *matrix, int n) {
    int j = threadIdx.x;

    extern __shared__ double localMatrix[];

    // Load matrix data into shared memory for faster access
    for (int i = 0; i < n; i++) {
        localMatrix[i * n + j] = matrix[i * n + j];
    }
    localMatrix[n * n + j] = 0; // sum column
    __syncthreads();

    // Perform Cholesky decomposition in parallel
    for (int i = 0; i < n; i++) {
        if (j < i) {
            localMatrix[i * n + j] = 0;
        } 
        if (j >= i) {
            double sum = 0; // local sum variable to reduce shared memory usage
            for (int k = 0; k < i; k++) {
                sum += localMatrix[k * n + i] * localMatrix[k * n + j];
            }
            if (i == j) {
                localMatrix[i * n + j] = sqrt(localMatrix[i * n + j] - sum);
            } else if (j > i) {
                localMatrix[i * n + j] = (localMatrix[i * n + j] - sum) / localMatrix[i * n + i];
            }
        }
    }
    __syncthreads();

    // Copy the result back to the global memory
    for (int i = 0; i < n; i++) {
        matrix[i * n + j] = localMatrix[i * n + j];
    }
}