#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use global index for computation
    c[i] = a[i] + b[i];
}