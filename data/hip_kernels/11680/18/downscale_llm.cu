#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    // Calculate global thread index
    long ii = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * (blockDim.x * gridDim.x);
    if (ii >= no_elements) return;
    
    // Pre-compute indices for the loop to reduce redundant computation
    int base_ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, 0, 0);
    int stride_d3 = d3 * scale_factor;

    // Unroll the loop for better efficiency
    for (int i = 0; i < scale_factor; i++) {
        int ipidx = base_ipidx + i * stride_d3;
        for (int j = 0; j < scale_factor; j++) {
            gradInput_data[ii] += gradOutput_data[ipidx + j];
        }
    }
}