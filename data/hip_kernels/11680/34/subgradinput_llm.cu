#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinput(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    
    // select input/output plane
    gradOutput = gradOutput + o * output_w * output_h;
    gradInput = gradInput + i * input_w * input_h;

    // get weight
    float the_weight = weight[k];

    // compute gradInput
    for (int yy = yy_start; yy < output_h; yy += blockDim.y * gridDim.y) {
        for (int xx = xx_start; xx < output_w; xx += blockDim.x) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput * the_weight;

            // Unroll the loops for kernel and weight computation
            #pragma unroll
            for (int ky = 0; ky < kH; ky++) {
                #pragma unroll
                for (int kx = 0; kx < kW; kx++)
                    ptr_gradInput[kx] += z;
                
                ptr_gradInput += input_w;
            }
        }
    }
}