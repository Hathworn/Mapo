#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate input data indices using integer division to avoid redundant calculations
    int time_index = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (time_index < output_w) {
        int gradInput_offset = blockIdx.x * input_w * input_n + time_index * input_n * dW;
        int gradOutput_offset = blockIdx.x * output_w * input_n + time_index * input_n;
        int indices_offset = blockIdx.x * output_w * input_n + time_index * input_n;

        float* gradInput_data = gradInput + gradInput_offset;
        float* gradOutput_data = gradOutput + gradOutput_offset;
        float* indices_data = indices + indices_offset;

        // Loop unrolling to improve memory access pattern and efficiency
        for (int feat = 0; feat < input_n; ++feat) {
            int idx = (int)indices_data[feat];
            gradInput_data[idx * input_n + feat] += gradOutput_data[feat];
        }
    }
}