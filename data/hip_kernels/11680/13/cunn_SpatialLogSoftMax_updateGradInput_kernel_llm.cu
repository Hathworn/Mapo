#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_SpatialLogSoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int classSize, int height, int width)
{
    int batchIndex = blockIdx.x;
    int index = threadIdx.x + blockDim.x * blockIdx.y; // Add blockIdx.y for better workload distribution

    // Loop with stride to handle more work items with a single kernel launch
    int stride = blockDim.x * gridDim.y;
    while (index < height * width) {
        int y = index / width;
        int x = index % width;

        if (y < height) { // Ensure valid y index range
            // Calculate output starting index in CUDA layout (B x H x W x C)
            int outputStartIndex = (height * width * classSize) * batchIndex + (width * classSize) * y + classSize * x;

            float sum = 0.0f;
            for (int i = 0; i < classSize; i++) {
                sum += gradOutput[outputStartIndex + i];
            }

            for (int i = 0; i < classSize; i++) {
                // Calculate input index in Torch layout (B x C x H x W)
                int inputIndex = (classSize * height * width) * batchIndex + (height * width) * i + width * y + x;
                gradInput[inputIndex] = gradOutput[outputStartIndex + i] - __expf(output[outputStartIndex + i]) * sum;
            }
        }
        index += stride; // Ensure all possible indices are covered by incrementing by stride
    }
}