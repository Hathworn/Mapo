#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // iterators
    int xx, yy;

    // output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y * gridDim.y;

    // select input/output plane
    gradOutput = gradOutput + o * output_w * output_h;
    gradInput = gradInput + i * input_w * input_h;

    // get weight
    float the_weight = weight[k];

    // compute gradInput
    for (yy = yy_start; yy < yy_end; yy += yy_step) {
        for (xx = xx_start; xx < xx_end; xx += xx_step) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput * the_weight;

            for (int idx = 0; idx < kH * kW; idx++) {
                int ky = idx / kW;  // Compute ky from linear index
                int kx = idx % kW;  // Compute kx from linear index

                // Use linear index for better memory access pattern
                atomicAdd(&(ptr_gradInput[kx + ky * input_w]), z);
            }
        }
    }
}