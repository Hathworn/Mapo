#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate the global thread index for the time dimension
    int time_index = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    
    if (time_index < output_w) {
        // Calculate pointers offset for current batch and time index
        float *gradInput_data = gradInput + blockIdx.x * input_w * input_n + time_index * input_n * dW;
        float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + time_index * input_n;
        float *indices_data = indices + blockIdx.x * output_w * input_n + time_index * input_n;
        
        // Unrolling loop to improve performance when input_n is a small and fixed known value
        for (int feat = 0; feat < input_n; ++feat) {
            // Use atomicAdd to accumulate gradient
            atomicAdd(&gradInput_data[(int)indices_data[feat] * input_n + feat], gradOutput_data[feat]);
        }
    }
}