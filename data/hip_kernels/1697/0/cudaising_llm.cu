#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaising(int* G, double* w, int* newG) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    double newSpin = 0.0;
    
    // Using shared memory for faster access to w
    __shared__ double shared_w[25];
    int tx = threadIdx.x;
    
    if (tx < 25) {
        shared_w[tx] = w[tx];
    }
    __syncthreads();
    
    // Improved loop access
    int halfDim = blockDim.x / 2;
    int xOffset = threadIdx.x + blockDim.x;
    int blockOffset = blockIdx.x + blockDim.x;
    
    for (int ii = -2; ii <= 2; ii++) {
        for (int jj = -2; jj <= 2; jj++) {
            // Pre-compute indexes for efficiency
            int wIndex = (jj + 2) + (ii + 2) * 5;
            int gIndex = ((jj + xOffset) % blockDim.x) + ((blockOffset + ii) % blockDim.x) * blockDim.x;
            newSpin += shared_w[wIndex] * G[gIndex];
        }
    }
    
    // Optimize conditions - avoiding multiple comparisons
    if (newSpin > 0.000001) {
        newG[index] = 1;
    } else if (newSpin < -0.000001) {
        newG[index] = -1;
    } else {
        newG[index] = G[index];
    }
}