#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaising(int* G, double* w, int* newG, int n, int workperthread) {
    int startingId = threadIdx.x * workperthread;

    // Shared memory for w and G
    __shared__ double tempW[25];  // 5 * 5 = 25
    __shared__ int tempG[521 * 5];  // (517 + 4) * 5

    // Optimized copy necessary elements from G into tempG using threads
    for (int j = -2; j <= n + 2; j++) {
        if (threadIdx.x < 5) {
            for (int i = -2; i <= 2; i++) {
                tempG[(j + 2) + (i + 2) * (n + 4)] = G[((j + n) % n) + ((blockIdx.x + i + n) % n) * n];
            }
        }
    }

    // Copy w into tempW
    if (threadIdx.x < 25) {
        tempW[threadIdx.x] = w[threadIdx.x];
    }
    __syncthreads();

    // Compute new spins
    for (int element = 0; element < workperthread; element++) {
        double newSpin = 0.0;

        // Calculate new spin value
        for (int ii = 0; ii < 5; ii++) {
            for (int jj = 0; jj < 5; jj++) {
                newSpin += tempW[jj + ii * 5] * tempG[startingId + element + jj + ii * (n + 4)];
            }
        }

        // Global index of the element
        int index = startingId + element + blockIdx.x * blockDim.x * workperthread;

        // Update the newG based on newSpin value
        newG[index] = (newSpin > 0.000001) ? 1 : (newSpin < -0.000001) ? -1 : G[index];
    }
    __syncthreads();
}