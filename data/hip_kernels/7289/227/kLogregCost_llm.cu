#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Use a bisection strategy for the correct probability calculation
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            // Reduce loop body size while counting max probabilities
            for (int i = 0; i < numOut; i++) {
                numMax += __float2int_rz(probs[i * numCases + tx] == maxp);
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}