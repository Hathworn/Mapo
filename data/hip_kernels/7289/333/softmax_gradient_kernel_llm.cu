#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    float tmp = 0;

    // Reduce inner products using loop unrolling for efficiency
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Reduce using a single warp for final reduction if within one warp
    if (idx < warpSize) {
        #pragma unroll
        for (int stride = warpSize / 2; stride > 0; stride /= 2) {
            tmp += __shfl_down_sync(0xffffffff, tmp, stride);
        }
    }

    // First thread writes the reduced result for all threads
    if (idx == 0) {
        reduction_buffer[0] = tmp;
    }
    __syncthreads();

    // Each thread computes its part of gradients.
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}