#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use blockIdx.x and threadIdx.x to ensure operation by only one thread
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        *accuracy /= N;
    }
}