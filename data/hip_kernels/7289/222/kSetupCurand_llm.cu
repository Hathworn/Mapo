#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate unique thread ID across the whole grid
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    // Initialize hiprand state - parallelized across threads
    hiprand_init(seed, tidx, 0, &state[tidx]);
}