#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define pi 3.14159265359f

__global__ void countCellOccKernal(unsigned int *d_hashArray, unsigned int *d_cellOccArray, int _hashTableSize, unsigned int _numPoints) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check index boundary and hash array limit
    if (idx < _numPoints) {
        unsigned int hashValue = d_hashArray[idx];
        if (hashValue < _hashTableSize) {
            // Atomically increment the occupancy count for the hash index
            atomicAdd(&(d_cellOccArray[hashValue]), 1);
        }
    }
}