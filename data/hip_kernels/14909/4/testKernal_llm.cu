#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void testKernal()
{
    // Use warp synchronization to reduce the number of printf calls
    if (threadIdx.x % warpSize == 0) {
        printf("warp leader thread number %d\n", threadIdx.x);
    }
}