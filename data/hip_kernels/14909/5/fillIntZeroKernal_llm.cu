#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fillIntZeroKernal(int *_bufferPtr, int size)
{
    // Use a loop to process multiple elements per thread to improve utilization
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    
    for (int i = idx; i < size; i += stride)
    {
        _bufferPtr[i] = 0;
    }
}