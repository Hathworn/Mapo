#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel2(int N, int *d_array) {
    // Use thread index to avoid sequential access
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        d_array[i] = i;
    }
}
```
