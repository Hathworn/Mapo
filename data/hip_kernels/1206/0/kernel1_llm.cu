#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Initialize array to 0
__global__ void kernel1(int N, int *d_array) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Compute global index
    if (i < N) {
        d_array[i] = 0; // Initialize to 0
    }
}
```
