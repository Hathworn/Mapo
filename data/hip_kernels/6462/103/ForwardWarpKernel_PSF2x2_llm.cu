#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Optimized computation of target pixel coordinates
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    int tx = static_cast<int>(floorf(cx));
    int ty = static_cast<int>(floorf(cy));
    float dx = cx - tx;
    float dy = cy - ty;

    // Ensure tx and ty are within bounds
    if (tx < 0 || tx >= w || ty < 0 || ty >= h) return;

    float value = src[image_row_offset + j];
    float weight;

    // Unrolled and simplified atomic operations
    weight = dx * dy;
    atomicAdd(dst + ty * image_stride + tx, value * weight);
    atomicAdd(normalization_factor + ty * image_stride + tx, weight);

    tx -= 1;
    weight = (1.0f - dx) * dy;
    if (tx >= 0) {
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    ty -= 1;
    weight = (1.0f - dx) * (1.0f - dy);
    if (ty >= 0) {
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    tx += 1;
    weight = dx * (1.0f - dy);
    if (tx < w) {
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}