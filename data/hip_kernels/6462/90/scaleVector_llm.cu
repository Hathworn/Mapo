#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Use grid and block dimensions to calculate global thread ID
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Launch threads only for valid positions
    if (pos < len) {
        d_res[pos] = d_src[pos] * scale;
    }
}