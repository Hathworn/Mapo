#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global thread index
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Use a single-dimensional thread index for coalesced memory access
    int globalIdx = i * w + j;

    // Ensure within bounds
    if (i < h && j < w) {
        image[globalIdx] = value;
    }
}