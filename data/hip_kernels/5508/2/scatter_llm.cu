#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scatter(unsigned int *d_in, unsigned int *d_index, unsigned int *d_out, unsigned int in_size) {
    // Use grid-stride loop for improved memory access pattern and handling larger arrays
    for (unsigned int index = threadIdx.x + blockDim.x * blockIdx.x; index < in_size; index += blockDim.x * gridDim.x) {
        d_out[d_index[index]] = d_in[index];
    }
}