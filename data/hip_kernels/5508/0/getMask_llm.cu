#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getMask(unsigned int *d_in, unsigned int *d_out, unsigned int in_size, unsigned int bit_shift, unsigned int One) {
    unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Check if index is within bounds
    if (index < in_size) {
        // Use ternary operator for brevity
        unsigned int bit = (d_in[index] & (1 << bit_shift)) ? 1 : 0;
        
        // Use ternary operator for clarity
        d_out[index] = One ? bit : 1 - bit;
    }
}
```

