#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x = ii / (d2 * d3);
    int y = (ii / d3) % d2;
    int z = (ii % d3) / scale_factor;
    int w = (ii % d3) % scale_factor;
    int new_d2 = d2 * scale_factor;
    int new_d3 = d3 * scale_factor;
    return (((x * d1 + y) * new_d2 + off_y * d3) * new_d3) + (z * new_d3) + (w * scale_factor + off_x);
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x = ii / (d2 * d3);
    int y = (ii / d3) % d2;
    int z = (ii % d3) / scale_factor;
    int w = (ii % d3) % scale_factor;
    int new_d2 = d2 / scale_factor;
    int new_d3 = d3 / scale_factor;
    return (((x * d1 + y) * new_d2 + z) * new_d3) + w;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x + (threadIdx.y + blockDim.y * blockIdx.y) * blockDim.x * gridDim.x;
    if (ii >= no_elements) return;

    // Unroll the loops for potential performance improvement
    #pragma unroll
    for (int i = 0; i < scale_factor; i++) {
        #pragma unroll
        for (int j = 0; j < scale_factor; j++) {
            int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
            gradInput_data[ii] += gradOutput_data[ipidx];
        }
    }
}