#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate the global thread ID for better memory access patterns
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;

    // Initialize the random number generator state for this thread
    hiprand_init(seed, tidx, 0, &state[tidx]);
}