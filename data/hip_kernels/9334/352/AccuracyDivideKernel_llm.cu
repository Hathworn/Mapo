#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel to handle multiple accuracy calculations
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global thread index
    if (idx == 0) { // Ensure only one thread performs the division
        *accuracy /= N;
    }
}