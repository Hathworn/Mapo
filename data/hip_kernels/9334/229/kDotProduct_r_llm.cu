#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    extern __shared__ float shmem[];

    // Calculate global thread index
    uint eidx = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    // Unrolling loop for better memory transaction efficiency
    // Each thread computes partial dot product
    for (uint idx = eidx; idx < numElements; idx += blockDim.x * gridDim.x) {
        sum += a[idx] * b[idx];
    }
    shmem[threadIdx.x] = sum;

    __syncthreads();

    // Perform reduction in shared memory
    for (uint stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Unrolled last warp
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    // Write block's result to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}