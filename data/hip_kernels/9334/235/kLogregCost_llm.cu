#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = static_cast<int>(labels[tx]);  // Use static_cast for type conversion
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Initialize numMax to 1 if the label probability equals max probability to prevent over-counting
        int numMax = (labelp == maxp) ? 1 : 0; 

        // Efficiently compute the number of maximum probabilities in parallel
        for (int i = 0; i < numOut; i++) {
            if (probs[i * numCases + tx] == maxp) {
                numMax++;
            }
        }

        // Calculate correct probability based on numMax
        correctProbs[tx] = 1.0f / static_cast<float>(numMax);
    }
}