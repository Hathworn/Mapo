#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    
    // Load and compute initial product in registers for efficiency
    float tmp = 0.0f;
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Reduce sum with loop unrolling (performance optimization)
    if (blockDim.x >= 1024 && idx < 512) reduction_buffer[idx] += reduction_buffer[idx + 512];
    __syncthreads();
    if (blockDim.x >= 512 && idx < 256) reduction_buffer[idx] += reduction_buffer[idx + 256];
    __syncthreads();
    if (blockDim.x >= 256 && idx < 128) reduction_buffer[idx] += reduction_buffer[idx + 128];
    __syncthreads();
    if (blockDim.x >= 128 && idx < 64) reduction_buffer[idx] += reduction_buffer[idx + 64];
    __syncthreads();

    if (idx < 32) {
        volatile float* vsmem = reduction_buffer; // volatile shared memory
        vsmem[idx] += vsmem[idx + 32];
        vsmem[idx] += vsmem[idx + 16];
        vsmem[idx] += vsmem[idx + 8];
        vsmem[idx] += vsmem[idx + 4];
        vsmem[idx] += vsmem[idx + 2];
        vsmem[idx] += vsmem[idx + 1];
    }
    __syncthreads();

    // Compute gradient using reduced result
    if (idx == 0) {
        reduction_buffer[0] = reduction_buffer[0]; // Store result
    }
    __syncthreads();

    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}