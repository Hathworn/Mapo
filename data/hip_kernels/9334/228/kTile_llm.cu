#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate the linear index of the current thread in the grid
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of threads
    const int numThreads = blockDim.x * gridDim.x;
    
    // Loop through target elements in steps of numThreads for coalesced memory access
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Avoid division/remainder for y/x calculation by using intrinsic functions to improve performance
        const uint y = __hip_divider(i, tgtWidth);
        const uint x = __hip_remainder(i, tgtWidth);
        
        // Calculate source indices using modulus for tiling
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Perform tiling operation
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}