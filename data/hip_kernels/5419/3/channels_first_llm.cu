#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void channels_first(float* input, float* rinput, int channels, int height, int width, int pad_size)
{
    // Compute global thread indices
    int n = blockIdx.x;
    int y = blockIdx.y;
    int x = blockIdx.z;
    int ch_off = threadIdx.x;
    
    // Precomputed dimensions for performance improvement
    int dimcyx = channels * height * width;
    int dimyx = height * width;
    
    int p_dimx = (width + 2 * pad_size);
    int p_dimy = (height + 2 * pad_size);
    int p_dimyxc = channels * p_dimy * p_dimx;
    int p_dimxc = p_dimx * channels;

    // Unrolling loop for better performance
    #pragma unroll
    for (int c = ch_off; c < channels; c += blockDim.x) {
        // Coalesced memory access and reduction of arithmetic inside the loop
        float value = input[n * dimcyx + c * dimyx + y * width + x];
        int output_index = n * p_dimyxc + 
                           (y + pad_size) * p_dimxc + 
                           (x + pad_size) * channels + c;
        rinput[output_index] = value;
    }
}