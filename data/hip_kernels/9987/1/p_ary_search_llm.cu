#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int get_index_to_check(int thread, int num_threads, int set_size, int offset) {
    return (((set_size + num_threads) / num_threads) * thread) + offset;
}

__global__ void p_ary_search(int search, int array_length, int *arr, int *ret_val) {
    const int num_threads = blockDim.x * gridDim.x;
    const int thread = blockIdx.x * blockDim.x + threadIdx.x;
    int set_size = array_length;

    if (thread == 0) {
        ret_val[0] = -1;
        ret_val[1] = 0;
    }
    __syncthreads(); // Synchronize after potential init

    while (set_size != 0) {
        int offset = ret_val[1];
        __syncthreads(); // Ensure all threads have the updated offset

        int index_to_check = get_index_to_check(thread, num_threads, set_size, offset);

        if (index_to_check < array_length) {
            int next_index_to_check = min(get_index_to_check(thread + 1, num_threads, set_size, offset), array_length - 1);

            if (search > arr[index_to_check] && search < arr[next_index_to_check]) {
                atomicExch(&ret_val[1], index_to_check); // Replace ret_val[1] safely
            } else if (search == arr[index_to_check]) {
                atomicExch(&ret_val[0], index_to_check); // Set found index atomically
            }
        }

        set_size = set_size / num_threads;
        __syncthreads(); // Sync up so no threads jump ahead
    }
}