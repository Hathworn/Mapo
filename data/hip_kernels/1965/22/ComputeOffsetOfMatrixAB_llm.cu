#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  // Calculate stride for grid-stride loop
    for (int32_t i = idx; i < N; i += stride) {
        output[blockIdx.x * N + i] = K_A_B - row_sum[blockIdx.x] - col_sum[i];
    }
}