#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel to normalize output
__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 

    // Use a loop unrolling technique to improve memory access pattern
    for (; idx < num_elements; idx += blockDim.x * gridDim.x) {
        // Directly update to_normalize array with computed indices
        int base_idx = idx * 3;
        to_normalize[base_idx] = batch_index;
        to_normalize[base_idx + 1] = class_index;
        to_normalize[base_idx + 2] = static_cast<int64_t>(original[idx]);
    }
}