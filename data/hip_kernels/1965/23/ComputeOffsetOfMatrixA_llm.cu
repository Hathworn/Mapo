#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride to handle large data sets
    for (int i = idx; i < N; i += gridDim.x * blockDim.x) {
        output[blockIdx.x * N + i] = -col_sum[i];
    }
}