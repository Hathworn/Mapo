#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Calculate the global index
    int32_t idx = blockIdx.x * N + threadIdx.x;
    
    // Unroll loops using grid-stride loop for better performance
    for (int32_t i = idx; i < (blockIdx.x + 1) * N; i += blockDim.x * gridDim.x) {
        output[i] = -row_sum[blockIdx.x];
    }
}