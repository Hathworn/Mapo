#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*******************************************************
* Copyright (c) 2015-2019, ArrayFire
* All rights reserved.
*
* This file is distributed under 3-clause BSD license.
* The complete license agreement can be obtained at:
* http://arrayfire.com/licenses/BSD-3-Clause
********************************************************/

#define USE_FORGE_CUDA_COPY_HELPERS

#define PI 3.14159265359

const unsigned DIMX      = 640;
const unsigned DIMY      = 480;
const float MINIMUM      = 1.0f;
const float MAXIMUM      = 20.f;
const float STEP         = 2.0f;

// Optimized kernel function
__global__ void pointGenKernel(float* points, float* dirs, int nelems, float minimum, float step) {
    // Calculate unique thread identifier for improved parallelism
    int id = blockDim.x * blockIdx.x + threadIdx.x + (blockDim.y * blockIdx.y + threadIdx.y) * nelems;
    
    if (id < nelems * nelems) { // Ensure within bounds
        int i = id % nelems;
        int j = id / nelems;
        
        float x = minimum + i * step;
        float y = minimum + j * step;
        
        points[2 * id + 0] = x;
        points[2 * id + 1] = y;
        
        float angle_multiplier = 2.0f * PI / 10.0f; // Precompute constant multiplier
        dirs[2 * id + 0] = sinf(angle_multiplier * x); // Reused multiplier
        dirs[2 * id + 1] = sinf(angle_multiplier * y); // Reused multiplier
    }
}