#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void colorsKernel(float* colors, hiprandState* states) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Combine array index calculations to minimize repetitive computation
    int color_index = 3 * id;

    // Fetch random state once and reuse
    hiprandState localState = states[id];

    // Generate random colors using the pre-fetched state
    colors[color_index] = hiprand_uniform(&localState);
    colors[color_index + 1] = hiprand_uniform(&localState);
    colors[color_index + 2] = hiprand_uniform(&localState);
}