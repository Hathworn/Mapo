#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void colorsKernel(float* colors, hiprandState* states) {
int id = blockIdx.x * blockDim.x + threadIdx.x;

colors[3 * id + 0] = hiprand_uniform(&states[id]);
colors[3 * id + 1] = hiprand_uniform(&states[id]);
colors[3 * id + 2] = hiprand_uniform(&states[id]);
}