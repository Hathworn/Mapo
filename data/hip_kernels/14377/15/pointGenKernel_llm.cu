#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pointGenKernel(float* points, float* dirs, int nBBS0, int nelems, float minimum, float step) {
    int blockId = blockIdx.y * (gridDim.x / nBBS0) + blockIdx.x;
    int k = blockId / nBBS0;
    int i = threadIdx.x + (blockIdx.x % nBBS0) * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure within bounds
    if (i < nelems && j < nelems && k < nelems) {
        // Precompute common expressions
        float iStep = i * step;
        float jStep = j * step;
        float kStep = k * step;
        
        float x = minimum + iStep;
        float y = minimum + jStep;
        float z = minimum + kStep;

        int id = i + j * nelems + k * nelems * nelems;

        // Set points and dirs
        points[3 * id + 0] = x;
        points[3 * id + 1] = y;
        points[3 * id + 2] = z;

        dirs[3 * id + 0] = x - 10.f;
        dirs[3 * id + 1] = y - 10.f;
        dirs[3 * id + 2] = z - 10.f;
    }
}