#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize: Using a shared state to reduce data access overhead
__global__ void histColorsKernel(float* histColors, hiprandState* states) {
    int bin = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load state once per thread
    hiprandState localState = states[bin];
    
    histColors[3 * bin + 0] = hiprand_uniform(&localState);
    histColors[3 * bin + 1] = hiprand_uniform(&localState);
    histColors[3 * bin + 2] = hiprand_uniform(&localState);
    
    // Save updated state back
    states[bin] = localState;
}