#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histColorsKernel(float* histColors, hiprandState* states) {
int bin = blockIdx.x * blockDim.x + threadIdx.x;

histColors[3 * bin + 0] = hiprand_uniform(&states[bin]);
histColors[3 * bin + 1] = hiprand_uniform(&states[bin]);
histColors[3 * bin + 2] = hiprand_uniform(&states[bin]);
}