#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mapKernel(float* out, int functionCode, float frange_start, float dx) {
    // Use shared memory for storing intermediate results
    extern __shared__ float shared[];

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    float x = frange_start + id * dx;
    float y;

    // Precompute indices for efficient memory access
    int outIdx = 2 * id;

    // Use if-else structure for potentially better performance on certain architectures
    if (functionCode == 0) {
        y = cos(x);
    } else if (functionCode == 1) {
        y = tan(x);
    } else {
        y = sin(x);
    }

    shared[threadIdx.x * 2] = x;
    shared[threadIdx.x * 2 + 1] = y;

    __syncthreads();  // Synchronize threads before writing to global memory

    out[outIdx] = shared[threadIdx.x * 2];
    out[outIdx + 1] = shared[threadIdx.x * 2 + 1];
}