#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randKernel(float* __restrict__ out, hiprandState* __restrict__ states, float min, float scale) {
    int id  = blockIdx.x * blockDim.x + threadIdx.x;
    // Prefetch state for potential better memory access
    hiprandState localState = states[id];
    // Use local state to generate random number
    float randomValue = hiprand_uniform(&localState) * scale + min;
    // Store result
    out[id] = randomValue;
    // Store the state back, if states can change per call
    states[id] = localState;
}