#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randKernel(float* out, hiprandState* states, float min, float scale) {
int id  = blockIdx.x * blockDim.x + threadIdx.x;
out[id] = hiprand_uniform(&states[id]) * scale + min;
}