#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel
__global__ void setupRandomKernel(hiprandState* states, unsigned long long seed) {
    // Calculate global thread index
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Initialize random state for the current thread
    if (tid < gridDim.x * blockDim.x) { // Boundary check to prevent out-of-bounds access
        hiprand_init(seed, tid, 0, &states[tid]);
    }
}