#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void genColorsKernel(float* colors, int nelems) {
    // Move constants to constant memory for better performance.
    __shared__ float AF_BLUE[3], AF_ORANGE[3];
    if (threadIdx.x == 0) {
        AF_BLUE[0] = 0.0588f; AF_BLUE[1] = 0.1137f; AF_BLUE[2] = 0.2745f;
        AF_ORANGE[0] = 0.8588f; AF_ORANGE[1] = 0.6137f; AF_ORANGE[2] = 0.0745f;
    }
    __syncthreads();

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Unroll the loop for better throughput.
    for (int idx = i; idx < nelems; idx += blockDim.x * gridDim.x) {
        float* color = (idx % 2 == 0) ? AF_ORANGE : AF_BLUE;
        // Minimize memory access by using local.
        colors[3 * idx + 0] = color[0];
        colors[3 * idx + 1] = color[1];
        colors[3 * idx + 2] = color[2];
    }
}