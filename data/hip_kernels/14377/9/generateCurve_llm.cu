#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void generateCurve(float t, float dx, float* out, const float ZMIN, const size_t ZSIZE) {
    // Calculate the global thread index
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    // Pre-calculate values used multiple times
    float z = ZMIN + offset * dx;
    float z_t = z * t + t;
    float reciprocal_z = 1.0f / z;
    float sin_t_scaled = 0.1f * sin(t);

    // Ensure thread index is within bounds
    if (offset < ZSIZE) {
        // Store results in the output array
        out[3 * offset]     = cos(z_t) * reciprocal_z;
        out[3 * offset + 1] = sin(z_t) * reciprocal_z;
        out[3 * offset + 2] = z + sin_t_scaled;
    }
}