#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_sinf(float* out, const size_t _data_size, int fnCode, const float _dx, const float _frange_start) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < _data_size) {
        float x = _frange_start + i * _dx;
        int idx = 2 * i;
        out[idx] = x;

        // Use a lookup table for function selection to enhance performance.
        float (*func_ptr)(float);
        switch (fnCode) {
            case 0: func_ptr = sinf; break;
            case 1: func_ptr = cosf; break;
            case 2: func_ptr = tanf; break;
            case 3: func_ptr = log10f; break;
            default: func_ptr = sinf; break; // Default case for safety.
        }
        out[idx + 1] = func_ptr(x);
    }
}