#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setupRandomKernel(hiprandState* states, unsigned long long seed) {
    // Calculate the global thread ID only once
    unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use initiate function to set up random state
    hiprand_init(seed, tid, 0, &states[tid]);
}