#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setupRandomKernel(hiprandState* states, unsigned long long seed) {
unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
hiprand_init(seed, tid, 0, &states[tid]);
}