#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shmem_reduce_kernel(float * d_out, const float * d_in)
{
    // Shared memory allocation
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // Load shared mem from global mem with boundary check
    if (myId < gridDim.x * blockDim.x) {
        sdata[tid] = d_in[myId];
    } else {
        sdata[tid] = 0.0f;  // Handle out of bounds
    }
    __syncthreads();  // Ensure entire block is loaded

    // Do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s && myId + s < gridDim.x * blockDim.x) {  // Boundary condition
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();  // Ensure all adds at one stage are done
    }

    // Only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}