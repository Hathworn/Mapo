#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK_X 16
#define THREADS_PER_BLOCK_Y 16

__global__ void tonemap( float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new, float  min_log_Y, float  max_log_Y, float  log_Y_range, int    num_bins, int    num_pixels_y, int    num_pixels_x )
{
    int image_index_2d_x = blockIdx.x * THREADS_PER_BLOCK_X + threadIdx.x;
    int image_index_2d_y = blockIdx.y * THREADS_PER_BLOCK_Y + threadIdx.y;
    int image_index_1d = image_index_2d_y * num_pixels_x + image_index_2d_x;

    // Ensure threads are within image bounds
    if (image_index_2d_x < num_pixels_x && image_index_2d_y < num_pixels_y)
    {
        float x = d_x[image_index_1d];
        float y = d_y[image_index_1d];
        float log_Y = d_log_Y[image_index_1d];
        
        // Pre-compute bin index outside of condition
        int bin_index = min(num_bins - 1, int((num_bins * (log_Y - min_log_Y)) / log_Y_range));
        float Y_new = d_cdf_norm[bin_index];

        // Use fused multiply-adds for potential optimization
        float inv_y = 1.0f / y;
        float X_new = x * (Y_new * inv_y);
        float Z_new = (1.0f - x - y) * (Y_new * inv_y);

        float r_new = fmaf(X_new, 3.2406f, fmaf(Y_new, -1.5372f, Z_new * -0.4986f));
        float g_new = fmaf(X_new, -0.9689f, fmaf(Y_new, 1.8758f, Z_new * 0.0415f));
        float b_new = fmaf(X_new, 0.0557f, fmaf(Y_new, -0.2040f, Z_new * 1.0570f));

        d_r_new[image_index_1d] = r_new;
        d_g_new[image_index_1d] = g_new;
        d_b_new[image_index_1d] = b_new;
    }
}