#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    // Load the normalization constant only once per block to save register usage.
    __shared__ float normalization_constant;
    if (threadIdx.x == 0) {
        normalization_constant = 1.f / d_input_cdf[n - 1];
    }
    __syncthreads(); // Ensure all threads have the normalization constant before proceeding.

    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_index_1d < n)
    {
        // Utilize shared memory for faster access to normalization constant.
        unsigned int input_value = d_input_cdf[global_index_1d];
        float output_value = input_value * normalization_constant;

        d_output_cdf[global_index_1d] = output_value;
    }
}