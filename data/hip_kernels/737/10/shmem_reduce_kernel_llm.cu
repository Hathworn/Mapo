#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shmem_reduce_kernel(float *d_out, const float *d_in)
{
    // Shared memory allocated in the kernel call
    extern __shared__ float sdata[];

    int myId = threadIdx.x + 2 * blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // Load shared memory from global memory
    sdata[tid] = d_in[myId] + d_in[myId + blockDim.x];
    __syncthreads(); // Ensure entire block is loaded

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s) // Only required threads perform work
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Ensure all adds at one stage are done
    }

    // Only thread 0 writes result for this block back to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}