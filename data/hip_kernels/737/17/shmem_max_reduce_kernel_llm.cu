#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shmem_max_reduce_kernel(float * d_max, const float * d_in) {
    // Allocate shared memory
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // Load shared mem from global mem
    sdata[tid] = d_in[myId];
    __syncthreads(); // Ensure entire block is loaded

    // Reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = max(sdata[tid], sdata[tid + s]);
        }
        __syncthreads(); // Ensure all max operations at one stage are done
    }

    // Thread 0 writes result for this block back to global mem
    if (tid == 0) {
        d_max[blockIdx.x] = sdata[0];
    }
}