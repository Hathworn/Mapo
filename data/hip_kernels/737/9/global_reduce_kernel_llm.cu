#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_reduce_kernel(float * d_out, float * d_in) {
    int myId = threadIdx.x + 2 * blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    // Optimization: use shared memory for reduction
    extern __shared__ float shared_data[];
    shared_data[tid] = d_in[myId] + d_in[myId + blockDim.x];
    __syncthreads();

    // Reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();  // Synchronize threads at each stage of reduction
    }

    // Only thread 0 writes result for this block back to global memory
    if (tid == 0) {
        d_out[blockIdx.x] = shared_data[0];
    }
}