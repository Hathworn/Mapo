#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_reduce_kernel(float *d_out, float *d_in) {
    extern __shared__ float sdata[];  // use shared memory for reduction
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // load input into shared memory
    sdata[tid] = d_in[myId];
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();  // make sure all adds at one stage are done!
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}