#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inc(int *array, size_t n) {
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    size_t stride = blockDim.x * gridDim.x; // Pre-compute stride

    for (; idx < n; idx += stride) { // Use for-loop
        array[idx]++;
    }
}