#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void row_sums(const float *A, float *sums, size_t ds) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x; // Create 1D thread index
    if (idx < ds) {
        float sum = 0.0f;
        const float *row = A + idx * ds; // Optimize access by calculating row start pointer
        for (size_t i = 0; i < ds; i++) {
            sum += row[i]; // Access elements using row pointer
        }
        sums[idx] = sum; // Store the sum in output
    }
}