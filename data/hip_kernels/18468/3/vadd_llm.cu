#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vadd(const float *A, const float *B, float *C, int ds) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a local variable for shared memory coalescing
    if (idx < ds) {
        float sum = A[idx] + B[idx];
        C[idx] = sum;
    }
}