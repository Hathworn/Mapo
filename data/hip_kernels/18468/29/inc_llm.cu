#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function by using shared memory for improved cache performance
__global__ void inc(int *array, size_t n) {
    extern __shared__ int sharedMem[];  // Declare shared memory
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    size_t stride = blockDim.x * gridDim.x; // Pre-calculate stride

    while (idx < n) {
        sharedMem[threadIdx.x] = array[idx]; // Load to shared memory
        __syncthreads(); // Ensure all loads to shared memory are complete

        sharedMem[threadIdx.x]++;  // Increment in shared memory
        __syncthreads(); // Ensure all increments are complete

        array[idx] = sharedMem[threadIdx.x]; // Store back to global memory
        idx += stride; // Move to next index
    }
}