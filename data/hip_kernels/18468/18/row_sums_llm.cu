#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void row_sums(const float *A, float *sums, size_t ds) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x; // create typical 1D thread index from built-in variables
    
    if (idx < ds) {
        float sum = 0.0f;
        
        // Use coalesced memory access by unrolling the loop
        int i = 0;
        for (; i + 3 < ds; i += 4) {
            sum += A[idx * ds + i];
            sum += A[idx * ds + i + 1];
            sum += A[idx * ds + i + 2];
            sum += A[idx * ds + i + 3];
        }

        // Handle remaining elements
        for (; i < ds; i++) {
            sum += A[idx * ds + i];
        }

        sums[idx] = sum;
    }
}