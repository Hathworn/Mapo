#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce(float *gdata, float *out, size_t n) {
    __shared__ float sdata[BLOCK_SIZE];
    int tid = threadIdx.x;
    int blockSize = blockDim.x;  // Cache blockDim.x to a local variable for reuse
    size_t idx = threadIdx.x + blockSize * blockIdx.x;

    float myMax = -FLT_MAX;  // Initialize with smallest possible float value

    // Grid stride loop to load data
    while (idx < n) {
        myMax = max(gdata[idx], myMax);
        idx += gridDim.x * blockSize;
    }
    sdata[tid] = myMax;  // Write to shared memory

    __syncthreads();
    // Parallel reduction in shared memory
    for (unsigned int s = blockSize / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = max(sdata[tid + s], sdata[tid]);
        }
        __syncthreads();
    }
    // Write result for this block to global memory
    if (tid == 0) out[blockIdx.x] = sdata[0];
}