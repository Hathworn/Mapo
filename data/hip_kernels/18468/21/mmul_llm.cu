#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mmul(const float *A, const float *B, float *C, int ds) {

    // Declare cache in shared memory
    __shared__ float As[block_size][block_size];
    __shared__ float Bs[block_size][block_size];

    int idx = threadIdx.x + blockDim.x * blockIdx.x; // Create thread x index
    int idy = threadIdx.y + blockDim.y * blockIdx.y; // Create thread y index

    float temp = 0; // Initialize temp outside of if-condition for later reuse

    // Use in-bound checks around computation for better warp efficiency
    if (idx < ds && idy < ds) {
        for (int i = 0; i < ds / block_size; i++) {

            // Load data into shared memory with boundary checks
            if (threadIdx.x < block_size && threadIdx.y < block_size) {
                As[threadIdx.y][threadIdx.x] = A[idy * ds + (i * block_size + threadIdx.x)];
                Bs[threadIdx.y][threadIdx.x] = B[(i * block_size + threadIdx.y) * ds + idx];
            }

            // Synchronize
            __syncthreads();

            // Perform calculations while caches are available
            for (int k = 0; k < block_size; k++)
                temp += As[threadIdx.y][k] * Bs[k][threadIdx.x]; // Dot product

            // Synchronize before next iteration
            __syncthreads();
        }

        // Write result to global memory if within bounds
        C[idy * ds + idx] = temp;
    }
}