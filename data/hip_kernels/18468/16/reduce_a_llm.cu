#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce_a(float *gdata, float *out) {
    __shared__ float sdata[BLOCK_SIZE];
    int tid = threadIdx.x;
    sdata[tid] = 0.0f;
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;

    // Unrolling loop to improve data loading
    while (idx < N) {
        sdata[tid] += gdata[idx];
        idx += gridDim.x * blockDim.x;
    }

    // Perform reduction in shared memory
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();  // Ensure all threads are synchronized before next iteration
    }
    
    // Use atomic operation for final result update
    if (tid == 0) atomicAdd(out, sdata[0]);
}