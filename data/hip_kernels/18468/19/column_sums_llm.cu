#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void column_sums(const float *A, float *sums, size_t ds) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x; // 1D thread index
    if (idx < ds) {
        float sum = 0.0f;
        // Use stride for better memory coalescing
        for (size_t i = idx; i < ds * ds; i += ds) {
            sum += A[i];
        }
        sums[idx] = sum;
    }
}