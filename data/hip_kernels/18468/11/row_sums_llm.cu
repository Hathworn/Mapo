#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void row_sums(const float *A, float *sums, size_t ds) {

    // Calculate row index
    int row = blockIdx.x;

    // Load row data into shared memory
    if (row < ds) {
        __shared__ float sdata[block_size];
        int tid = threadIdx.x;
        sdata[tid] = 0.0f;
        size_t tidx = tid;

        // Striding over the row to load data into shared memory
        while (tidx < ds) {
            sdata[tid] += A[row * ds + tidx];
            tidx += blockDim.x;
        }

        // Use parallel reduction to sum elements in shared memory
        __syncthreads();
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                sdata[tid] += sdata[tid + s];
            }
            __syncthreads();
        }

        // Store the result back to global memory
        if (tid == 0) {
            sums[row] = sdata[0];
        }
    }
}