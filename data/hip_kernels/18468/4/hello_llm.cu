#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello() {
    // Precompute block index and thread index in registers for faster access
    unsigned int blockIndex = blockIdx.x;
    unsigned int threadIndex = threadIdx.x;

    // Minimize printf calls for performance boost
    printf("Hello from block: %u, thread: %u\n", blockIndex, threadIndex);
}