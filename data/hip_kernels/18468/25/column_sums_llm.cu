#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void column_sums(const float *A, float *sums, size_t ds) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x; // create typical 1D thread index from built-in variables
    if (idx < ds) {
        float sum = 0.0f;
        A += idx; // Pointer offset to start at column
        // Unroll loop to improve performance
        for (size_t i = 0; i < ds; i += 4) {
            sum += A[ds * i];
            if (i + 1 < ds) sum += A[ds * (i + 1)];
            if (i + 2 < ds) sum += A[ds * (i + 2)];
            if (i + 3 < ds) sum += A[ds * (i + 3)];
        }
        sums[idx] = sum;
    }
}