#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// error checking macro
__global__ void mmul(const float *A, const float *B, float *C, int ds) {
    // Thread coordinates
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;
    
    // Boundary check
    if (idx < ds && idy < ds) {
        // Initialize accumulation
        float temp = 0.0f;
        
        // Coalesced access for A and B
        for (int i = 0; i < ds; i++) {
            temp += A[idy * ds + i] * B[i * ds + idx];
        }
        
        // Write result back to C
        C[idy * ds + idx] = temp;
    }
}