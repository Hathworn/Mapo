#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomic_red(const float *gdata, float *out){
    // Calculate global thread index
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Use grid-stride loop to improve memory coalescing
    for (size_t i = idx; i < N; i += gridDim.x * blockDim.x) {
        atomicAdd(out, gdata[i]);
    }
}