#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved global kernel function with better access pattern
__global__ void vadd(const float *A, const float *B, float *C, int ds) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    for (; idx < ds; idx += stride) {
        C[idx] = A[idx] + B[idx];
    }
}