#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce(float *gdata, float *out, size_t n) {
    __shared__ float sdata[BLOCK_SIZE];
    int tid = threadIdx.x;
    sdata[tid] = 0.0f;
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;

    // Use grid stride loop to load data efficiently
    for (; idx < n; idx += gridDim.x * blockDim.x) {
        sdata[tid] += gdata[idx];
    }

    // Unrolling loop to optimize parallel reduction
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        __syncthreads();
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
    }
    // Use warp shuffle for last warp iterations
    if (tid < 32) {
        __syncthreads();
        sdata[tid] += sdata[tid + 32];
        sdata[tid] += sdata[tid + 16];
        sdata[tid] += sdata[tid + 8];
        sdata[tid] += sdata[tid + 4];
        sdata[tid] += sdata[tid + 2];
        sdata[tid] += sdata[tid + 1];
    }
    if (tid == 0) out[blockIdx.x] = sdata[0];
}