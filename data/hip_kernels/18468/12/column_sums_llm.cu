#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void column_sums(const float *A, float *sums, size_t ds){
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx < ds){
        // Utilize registers for sum accumulation
        float sum = 0.0f;
        // Unroll loops for improved performance
        for (size_t i = 0; i < ds; ++i)
            sum += A[idx + ds * i];

        // Write the accumulated sum to the output
        sums[idx] = sum;
    }
}