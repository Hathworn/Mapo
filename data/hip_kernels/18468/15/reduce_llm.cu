#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce(float *gdata, float *out) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    float sum = 0.0f;

    // Grid-stride loop for loading data
    while (idx < N) {
        sum += gdata[idx];
        idx += gridDim.x * blockDim.x;
    }

    // Store partial sum in shared memory
    sdata[tid] = sum;
    __syncthreads();

    // Perform parallel reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result from first thread of this block
    if (tid == 0) out[blockIdx.x] = sdata[0];
}