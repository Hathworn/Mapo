#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
152096 - William Matheus
Friendly Numbers
Programacao Paralela e Distribuida
CUDA - 2019/2 - UPF
Programa 2 - Kernel
*/

__global__ void sum(long int* device_num, long int* device_den, long int* device_vet, int size, int x)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + x;
    
    // Use shared memory for reductions
    extern __shared__ long int shared_mem[];
    long int *shared_num = shared_mem;
    long int *shared_den = shared_mem + blockDim.x;

    if (i < size) {
        shared_num[threadIdx.x] = device_num[i];
        shared_den[threadIdx.x] = device_den[i];

        __syncthreads();

        // Local computations within the block
        for (int j = threadIdx.x + 1; j < blockDim.x; j++) {
            int idx = blockIdx.x * blockDim.x + j + x;
            if (idx < size) {
                if ((shared_num[threadIdx.x] == device_num[idx]) && (shared_den[threadIdx.x] == device_den[idx])) {
                    atomicAdd(&device_vet[i], 1);
                }
            }
        }
    }
}