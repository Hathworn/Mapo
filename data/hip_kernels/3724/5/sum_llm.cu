#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(long int* device_num, long int* device_den, long int* device_vet, int size, int x)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + x;
    int j;

    // Check bounds to avoid unnecessary computations.
    if (i < size) {
        // Use shared memory for repeated accesses within the same block.
        __shared__ long int shared_num[1024];
        __shared__ long int shared_den[1024];

        // Load data into shared memory.
        shared_num[threadIdx.x] = device_num[i];
        shared_den[threadIdx.x] = device_den[i];
        __syncthreads();

        for (j = i + 1; j < size; j++) {
            // Minimize global memory accesses by using shared memory.
            if ((shared_num[threadIdx.x] == device_num[j]) && (shared_den[threadIdx.x] == device_den[j])) {
                atomicAdd(&device_vet[i], 1); // Use atomic operation for thread safety.
            }
        }
    }
}