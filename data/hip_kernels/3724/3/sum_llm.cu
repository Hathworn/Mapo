#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum(long int* device_num, long int* device_den, long int* device_vet, int size, int x)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + x;
    int j;

    // Use shared memory for device_num and device_den to reduce global memory access
    extern __shared__ long int shared_mem[];
    long int* shared_num = shared_mem;
    long int* shared_den = shared_mem + size;

    if (i < size) {
        shared_num[i] = device_num[i];
        shared_den[i] = device_den[i];
    }
    __syncthreads();

    if (i < size) {
        for (j = i + 1; j < size; j++) {
            if ((shared_num[i] == shared_num[j]) && (shared_den[i] == shared_den[j]))
                atomicAdd(&device_vet[i], 1); // Use atomic operation to avoid race conditions
        }
    }
}