#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DEBUG false
#define DEBUG_OUTPUT false
#define DEBUG_DELTA_K false
#define DEBUGNET false
#define DEBUG_TIMEING true
#define index(i,j,ld) (((j)*(ld))+(i))

int numBlocks = 1;
int blockSize = 256;

using namespace std;

/*
*  Print Matrix on host
*/
__global__ void addConstant(float* input, float constant, int num_elements) {
    // Calculate the global thread ID
    const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the thread ID is within range
    if (tid < num_elements) {
        // Add the constant to the input element
        input[tid] += constant;
    }
}