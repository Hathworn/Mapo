#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DEBUG false
#define DEBUG_OUTPUT false
#define DEBUG_DELTA_K false
#define DEBUGNET false
#define DEBUG_TIMEING true
#define index(i,j,ld) (((j)*(ld))+(i))

int numBlocks = 1;
int blockSize = 256;

using namespace std;

/*
*  Print Matrix on host
*/
__global__ void squaredError(float* predicted_values, float* actual_values, float* results, int num_elements){
    // Calculate the global thread index
    const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if within array bounds
    if (tid < num_elements) {
        // Calculate squared error
        float diff = actual_values[tid] - predicted_values[tid];
        results[tid] = diff * diff;
    }
}