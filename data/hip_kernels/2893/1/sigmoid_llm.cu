#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DEBUG false
#define DEBUG_OUTPUT false
#define DEBUG_DELTA_K false
#define DEBUGNET false
#define DEBUG_TIMEING true
#define index(i,j,ld) (((j)*(ld))+(i))

int numBlocks = 1;
int blockSize = 256;

using namespace std;

/*
*  Print Matrix on host
*/
__global__ void sigmoid(float* input, int num_elements) {
    // Calculate global thread ID
    const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Ensure thread is within bounds
    if (tid < num_elements) {
        // Use math function optimization
        float value = __fdividef(1.0f, (1.0f + expf(-input[tid])));
        input[tid] = value;
    }
}