#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiply(float *A, float *B, float *C, int N, int threads_num) {
    extern __shared__ float sharedMem[];  // Use dynamic shared memory
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int a = N * N * thread_id / threads_num;
    int b = (thread_id == threads_num - 1) ? (N * N) : (N * N * (thread_id + 1) / threads_num);

    // Cache part of matrix B to shared memory
    for (int i = threadIdx.x; i < N; i += blockDim.x) {
        sharedMem[i] = B[i];
    }
    __syncthreads();

    for (int pos = a; pos < b; ++pos) {
        float tmp = 0;
        int row = pos / N;
        int col = pos % N;
        for (int k = 0; k < N; ++k) {
            tmp += A[row * N + k] * sharedMem[k * N + col]; // Use shared memory for B
        }
        C[pos] = tmp;
    }
}