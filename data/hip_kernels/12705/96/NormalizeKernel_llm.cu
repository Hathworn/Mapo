#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Early return if out of the processing bounds
    if (i >= h || j >= w) return;

    // Calculate position index
    int pos = i * s + j;

    // Load normalization factor
    float scale = normalization_factor[pos];

    // Compute inverse scale safely
    float invScale = (scale != 0.0f) ? (1.0f / scale) : 1.0f;

    // Scale image at current position
    image[pos] *= invScale;
}