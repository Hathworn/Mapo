#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for better performance if enough shared memory is available
    __shared__ short2 loc_shared[256]; // Adjust size based on available memory and block size

    if (ptidx < npoints) {
        loc_shared[threadIdx.x] = loc_[ptidx]; // Load data into shared memory
        __syncthreads();

        short2 loc = loc_shared[threadIdx.x]; // Read from shared memory
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}