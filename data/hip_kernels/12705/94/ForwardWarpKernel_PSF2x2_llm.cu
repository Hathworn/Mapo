#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate the offset and pixel indices
    float cx = u[flow_row_offset + j] * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + static_cast<float>(i) + 1.0f;

    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);

    float value = src[image_row_offset + j];

    // Pre-compute the weights
    float weight_br = dx * dy;
    float weight_bl = (1.0f - dx) * dy;
    float weight_tl = (1.0f - dx) * (1.0f - dy);
    float weight_tr = dx * (1.0f - dy);

    // Fill pixel containing bottom right corner
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        _atomicAdd(dst + ty * image_stride + tx, value * weight_br);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight_br);
    }

    tx -= 1;
    // Fill pixel containing bottom left corner
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        _atomicAdd(dst + ty * image_stride + tx, value * weight_bl);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight_bl);
    }

    ty -= 1;
    // Fill pixel containing upper left corner
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        _atomicAdd(dst + ty * image_stride + tx, value * weight_tl);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight_tl);
    }

    tx += 1;
    // Fill pixel containing upper right corner
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        _atomicAdd(dst + ty * image_stride + tx, value * weight_tr);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight_tr);
    }
}