#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void showProgress(char* trystr, int tam, unsigned long long int id)
{
    int n = 0;

    for (int i = 0; i < tam; i++)
    {
        if ('!' != trystr[i])
        {
            n = 1;
            break;
        }
    }

    if (id % 1000000000 == 0) printf("\n----- Id = %lld\n", id);

    if (n == 0) printf("\n***** %lld, Tam = %d -> ┤%s├\n", id, tam, trystr);
}

__device__ char * id2str(unsigned long long int n)
{
    int b = 96, r, asciChar, count = 0;
    char * a, * res;

    a = (char*) malloc(100);

    do
    {
        r = n % b;
        asciChar = 32 + r;
        a[count] = asciChar;
        count++;
        n = n / b;
    }
    while (n != 0);

    res = (char*) malloc(count + 1);

    for (int i = count - 1; i >= 0; --i) res[count - i - 1] = a[i];

    free(a);

    res[count] = '\0';

    return res;
}

__device__ __host__  int shc(char* ca, int tam)
{
    int h = 0;

    for (int i = 0; i < tam; i++)
    {
        h = 31 * h + ca[i];
    }

    return h;
}

__global__ void findcollisions(int hash, unsigned long long int strLength)
{
    unsigned long long int id = blockDim.x * blockIdx.x + threadIdx.x + strLength;
    char *trystr = id2str(id);
    int hc;

    // Optimize: Use `strlen` for readability and efficiency
    int tam = strlen(trystr); 

    showProgress(trystr, tam, id);

    hc = shc(trystr, tam);

    if (hc == hash)
        printf("Collision found for string ┤%s├. Hashcode %d\n", trystr, hc);

    if (trystr[tam - 1] == ' ')
    {
        char * reverse = (char *) malloc(tam + 1);

        // Optimize: Use `memcpy` and reverse loop
        for (int i = 0; i < tam; ++i)
            reverse[i] = trystr[tam - 1 - i];

        reverse[tam] = '\0';

        hc = shc(reverse, tam);

        if (hc == hash)
            printf("Collision found for string ┤%s├. Hashcode %d\n", reverse, hc);

        free(reverse);
    }
    free(trystr);
}