#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexCopy( float *res, float *src, long* res_stride, float *index, long res_nDim, int dim, long idx_size, long src_size, long size_dim )
{
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    long flat_size = src_size / idx_size;

    // Use shared memory to store res_stride for faster access
    extern __shared__ long shared_res_stride[];
    if (threadIdx.x < res_nDim) {
        shared_res_stride[threadIdx.x] = res_stride[threadIdx.x];
    }
    __syncthreads();

    if (thread_idx < flat_size)
    {
        for (int i = 0; i < idx_size; i++)
        {
            int leftover = thread_idx;
            int targetIdx = 0;
            int resIdx = 0;
            for (int d = 0; d < res_nDim; d++)
            {
                long stride_d = (d == dim) ? size_dim : shared_res_stride[d];
                long coeff = leftover / stride_d;
                leftover -= coeff * stride_d;
                if (d < dim)
                {
                    targetIdx += coeff * stride_d * idx_size;
                    resIdx += coeff * shared_res_stride[d];
                }
                else if (d > dim)
                {
                    targetIdx += coeff * shared_res_stride[d];
                    resIdx += coeff * shared_res_stride[d];
                }
            }
            long idx_offset = ((long)(index[i]) - 1) * shared_res_stride[dim];
            res[resIdx + idx_offset] = src[targetIdx + i * shared_res_stride[dim]];
        }
    }
}