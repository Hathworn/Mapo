#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;
        float midVal = dist[mid];
        
        // Check mid value against threshold
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    // Handle edge case outside range
    if (start == size) {
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithoutReplacement(hiprandStateMtgp32* state, int totalSamples, int sample, float* dest, long distributions, int categories, float* origDist, float* normDistPrefixSum) {
    // Efficient loop unrolling and memory access
    for (long curDistBase = blockIdx.x * blockDim.y; curDistBase < distributions; curDistBase += gridDim.x * blockDim.y) {
        long curDist = curDistBase + threadIdx.y;
        
        // Uniform generation outside critical section
        __shared__ float r;
        if (threadIdx.x == 0) {
            if (curDist < distributions) {
                r = hiprand_uniform(&state[blockIdx.x]);
            }
        }
        __syncthreads();

        if (threadIdx.x == 0 && curDist < distributions) {
            // Perform binary search on the cumulative distribution
            int choice = binarySearchForMultinomial(normDistPrefixSum + curDist * categories, categories, r);

            // Assign sample to output with correction for indexing
            dest[curDist * totalSamples + sample] = (float) choice + 1.0f;

            // Zero out the used probability
            origDist[curDist * categories + choice] = 0.0f;
        }
    }
}