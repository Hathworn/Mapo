#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexSelect_contiguous(float *tensor, float *src, long stride, float *index, long idxSize) {
    // Constants for vectorized processing
    const int VT = 4;
    const int WARP_SIZE = 32;
    const int MAX_DIM_SIZE = 65535;

    // Calculate starting position in the index array per thread block
    int idx = blockIdx.x * blockDim.y + threadIdx.y;
    while (idx < idxSize) {
        // Calculate starting position in tensor source per thread block
        int startIdx = threadIdx.x + blockIdx.y * VT * WARP_SIZE;
        while (startIdx < stride) {
            // Compute source and target indices
            const long srcIdx = ((long)index[idx] - 1) * stride;
            const long targetIdx = idx * stride;
            
            // Unroll loop to improve memory coalescing
            #pragma unroll
            for (int i = 0; i < VT; i++) {
                const int featureIdx = startIdx + i * WARP_SIZE;
                if (featureIdx < stride) {
                    tensor[targetIdx + featureIdx] = src[srcIdx + featureIdx];
                }
            }
            
            // Step to the next section in the stride
            startIdx += VT * WARP_SIZE * MAX_DIM_SIZE;
        }
        
        // Step to the next section in the idxSize
        idx += blockDim.y * MAX_DIM_SIZE;
    }
}