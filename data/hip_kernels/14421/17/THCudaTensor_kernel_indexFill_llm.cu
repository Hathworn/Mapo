#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void THCudaTensor_kernel_indexFill( float *tensor, long* stride, float *index, long src_nDim, int dim, long idx_size, long tensor_size, long size_dim, float val )
{
    // Calculate the global thread index using a typical 3D grid and block approach
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    long flat_size = tensor_size / idx_size;
    
    // Check if the thread index is within bounds
    if (thread_idx < flat_size)
    {
        // Precompute and store index multipliers to avoid repetitive calculation
        long idxStride = stride[dim];
        
        for (int i = 0; i < idx_size; i++)
        {
            int leftover = thread_idx;
            int srcIdx = 0;

            // Loop through dimensions and calculate the index in tensor
            for (int d = 0; d < src_nDim; d++)
            {
                long step = (d < dim) ? stride[d] / size_dim : stride[d];
                long coeff = leftover / step;
                leftover -= coeff * step;
                srcIdx += coeff * stride[d];
            }
            
            // Update tensor value for the calculated index
            tensor[srcIdx + (long)(index[i] - 1) * idxStride] = val;
        }
    }
}