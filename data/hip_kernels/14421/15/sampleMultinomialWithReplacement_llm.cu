#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;
        
        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithReplacement(hiprandStateMtgp32* state, int totalSamples, float* dest, long distributions, int categories, float* normDistPrefixSum) {
    for (long curDist = blockIdx.x; curDist < distributions; curDist += gridDim.x) {
        for (int sampleBase = 0; sampleBase < totalSamples; sampleBase += blockDim.y) {
            int sample = sampleBase + threadIdx.y;

            float r = hiprand_uniform(&state[blockIdx.x]);

            // Parallelize binary search using warp-level synchronization
            if (sample < totalSamples) {
                __shared__ int sharedChoice[32];  // Use shared memory for warp-level reduction
                int choice = binarySearchForMultinomial(
                    normDistPrefixSum + curDist * categories,
                    categories,
                    r
                );
                sharedChoice[threadIdx.y] = choice;  // Store result in shared memory

                __syncthreads();  // Synchronize threads within the warp
                if (threadIdx.x == 0) {  // First thread in the warp writes the result
                    dest[curDist * totalSamples + sample] = (float) sharedChoice[threadIdx.y] + 1.0f;
                }
            }
        }
    }
}