#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn){
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < dim_mn) {
        double out = 0.0;
        #pragma unroll // Unroll loop for better performance
        for (long j = 0; j < dim_mn; j++) {
            out += matrix[i * dim_mn + j] * vector_in[j];
        }
        vector_out[i] = out;
    }
}