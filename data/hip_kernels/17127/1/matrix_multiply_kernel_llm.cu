#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    // Use shared memory for vector_in to reduce global memory access
    extern __shared__ double shared_vector[];
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Load vector_in into shared memory
    if (threadIdx.x < dim_mn) {
        shared_vector[threadIdx.x] = vector_in[threadIdx.x];
    }
    __syncthreads();
    
    if (i < dim_mn) {
        double out = 0.0;
        for (long j = 0; j < dim_mn; j++) {
            out += matrix[i * dim_mn + j] * shared_vector[j];
        }
        vector_out[i] = out;
    }
}