#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLKX 32
#define BLKY 32

hipStream_t gstream;

__global__ void gpuWork(double *g, double *h, double *error, int M, int nbLines) {
    // Calculate global indices considering the halo region
    long idX = threadIdx.x + blockIdx.x * blockDim.x + 1;
    long idY = threadIdx.y + blockIdx.y * blockDim.y + 1;
    long tidX = threadIdx.x + blockIdx.x * blockDim.x;
    long tidY = threadIdx.y + blockIdx.y * blockDim.y;
    long xSize = M + 2;

    // Utilize shared memory for error calculation
    __shared__ double errors[BLKX * BLKY];

    // Initialize to zero
    errors[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;

    // Check bounds
    if (tidX < M && tidY < nbLines) {
        // Calculate new value
        double temp = 0.25 * (h[(idY - 1) * xSize + idX] +
                              h[(idY + 1) * xSize + idX] +
                              h[(idY * xSize) + idX - 1] +
                              h[(idY * xSize) + idX + 1]);
        // Store the result and calculate error
        g[(idY * xSize) + idX] = temp;
        errors[threadIdx.y * blockDim.x + threadIdx.x] = fabs(temp - h[(idY * xSize) + idX]);
    }

    // Synchronize threads within the block
    __syncthreads();

    // Reduce errors within the block
    for (unsigned long s = (blockDim.x * blockDim.y) / 2; s > 0; s >>= 1) {
        if (threadIdx.y * blockDim.x + threadIdx.x < s) {
            errors[threadIdx.y * blockDim.x + threadIdx.x] = 
                fmax(errors[threadIdx.y * blockDim.x + threadIdx.x],
                     errors[threadIdx.y * blockDim.x + threadIdx.x + s]);
        }
        __syncthreads();
    }

    // Write maximum error per block to global memory
    if (threadIdx.y * blockDim.x + threadIdx.x == 0) {
        int id = blockIdx.y * gridDim.x + blockIdx.x;
        error[id] = errors[0];
    }
}