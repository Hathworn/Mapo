#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLKX 32
#define BLKY 32

hipStream_t gstream;

__global__ void initData(int nbLines, int M, double *h, double *g)
{
    // Compute unique thread index
    long idX = threadIdx.x + blockIdx.x * blockDim.x;
    long totalSize = nbLines * M;

    // Check if the thread is within bounds
    if (idX >= totalSize) return;

    // Initialize arrays
    h[idX] = 0.0L;
    g[idX] = 0.0L;
    
    // Apply condition and update arrays
    if (idX >= M + 1 && idX < 2 * M - 1) {
        h[idX] = 100.0;
        g[idX] = 100.0;
    }
}