#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved global kernel function
__global__ void kernel(int* d_vec, int n) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // Account for global thread id

    if (tid < n) {
        int i = d_vec[tid];
        d_vec[tid] = i > 5 ? -i : i;
    }
}