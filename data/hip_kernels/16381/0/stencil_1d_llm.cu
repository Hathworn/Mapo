#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define RADIUS        3
#define BLOCK_SIZE    256
#define NUM_ELEMENTS  (4096*2)

__global__ void stencil_1d(int *in, int *out)
{
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + blockIdx.x * blockDim.x + RADIUS;
    int lindex = threadIdx.x + RADIUS;

    // Load the main body into shared memory
    temp[lindex] = in[gindex];
    
    // Load halo elements
    if (threadIdx.x < RADIUS){
        temp[lindex - RADIUS] = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    __syncthreads();

    // Compute stencil operation
    int result = 0;
    #pragma unroll  // Unroll loop for better performance
    for (int offset = -RADIUS; offset <= RADIUS; ++offset)
        result += temp[lindex + offset];

    out[gindex - RADIUS] = result;
}