#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
    // Compute global index
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Check to ensure the index does not exceed the array bounds
    if (index < blockDim.x * gridDim.x) {
        c[index] = a[index] + b[index];
    }
}