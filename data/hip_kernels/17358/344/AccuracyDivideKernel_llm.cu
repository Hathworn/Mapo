#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
  // Use atomic operation for thread-safe division
  atomicExch(accuracy, *accuracy / N);
}