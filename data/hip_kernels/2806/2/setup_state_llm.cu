#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Optimize by setting unique sequence numbers for each thread
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}