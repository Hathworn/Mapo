#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Precompute constants to minimize repeated calculations.
    const int stride = blockDim.x * gridDim.x;
    const int offset_batch_index = batch_index * 3;
    const int offset_class_index = class_index * 3 + 1;

    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_elements; idx += stride) {
        int base_idx = idx * 3;
        to_normalize[base_idx] = offset_batch_index;
        to_normalize[base_idx + 1] = offset_class_index;
        to_normalize[base_idx + 2] = static_cast<int64_t>(original[idx]);
    }
}