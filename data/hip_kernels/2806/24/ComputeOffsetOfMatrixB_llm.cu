#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Cache row_sum in a local variable
    int32_t row_sum_value = row_sum[blockIdx.x];
    
    // Use grid stride loop for better memory access
    for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
        *(output + blockIdx.x * N + i) = -row_sum_value;
    }
}