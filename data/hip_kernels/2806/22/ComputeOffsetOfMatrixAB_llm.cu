#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use 1D grid-stride loop to ensure all elements are processed
    for (int i = idx; i < blockDim.x * gridDim.x && i < N; i += blockDim.x * gridDim.x) {
        output[i] = K_A_B - row_sum[blockIdx.x] - col_sum[i];
    }
}