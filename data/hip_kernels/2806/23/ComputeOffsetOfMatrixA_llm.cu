#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Optimize using shared memory and reduce global memory accesses
    extern __shared__ int32_t shared_col_sum[];
    
    for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
        shared_col_sum[i] = col_sum[i];
    }
    __syncthreads();

    for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
        output[blockIdx.x * N + i] = -shared_col_sum[i];
    }
}