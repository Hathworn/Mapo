#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kEqualsScalar(float* __restrict__ mat, float val, float* __restrict__ target, unsigned int len) {
    // Calculate the thread's unique index in the grid
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Utilize shared memory to enhance memory access speed (assuming small block sizes and data fits)
    extern __shared__ float shared_mat[];
    
    // Check if index is within bounds
    if (idx < len) {
        // Load mat into shared memory
        shared_mat[threadIdx.x] = mat[idx];
        __syncthreads();
        
        // Perform computation
        target[idx] = shared_mat[threadIdx.x] == val;
    }
}