#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function
__global__ void kDivideScalar(float* mat, float alpha, float* dest, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Replace numThreads with stride for better readability and consistency
    for (unsigned int i = idx; i < len; i += stride) {
        dest[i] = mat[i] / alpha;
    }
}