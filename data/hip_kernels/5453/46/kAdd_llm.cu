#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd(float* __restrict__ a, float* __restrict__ b, float* __restrict__ dest, unsigned int numEls) {
    // Calculate the unique thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a block-wide stride to allow unrolled loop for better memory access pattern
    if(idx < numEls) {
        dest[idx] = a[idx] + b[idx];
    }
}