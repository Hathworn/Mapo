#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSelectRows(float* source, float* target, float* indices, int nRowIs, int nCols, int nSourceRows){
    __shared__ int sourceRowIndices[32];
    const int startTargetRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs-startTargetRowI);

    // Load row indices into shared memory
    if (tid < localNRowIs) {
        int index = int(indices[startTargetRowI + tid]);
        index = index < 0 ? index + nSourceRows : index;
        sourceRowIndices[tid] = (index < 0 || index >= nSourceRows) ? -1 : index;
    }
    __syncthreads();

    // Efficiently copy rows using optimized memory accesses
    for (int i = 0; i < localNRowIs; i++) {
        int sourceRowI = sourceRowIndices[i];
        if (sourceRowI != -1) {
            for (int colI = tid; colI < nCols; colI += 32) {
                target[(startTargetRowI + i) * nCols + colI] = source[sourceRowI * nCols + colI];
            }
        } else if (tid == 0) {  // handle NaN for invalid indices once per row
            for (int colI = 0; colI < nCols; colI++) {
                target[(startTargetRowI + i) * nCols + colI] = NAN;
            }
        }
    }
}