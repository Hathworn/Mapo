#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kGreaterThan(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by using a single loop iteration per thread
    if (idx < len) {
        target[idx] = mat1[idx] > mat2[idx];
    }
}