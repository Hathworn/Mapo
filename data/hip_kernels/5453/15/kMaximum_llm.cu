#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMaximum(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x; // Improve variable naming

    // Use __restrict__ to suggest that there is no aliasing
    float* __restrict__ m1 = mat1;
    float* __restrict__ m2 = mat2;
    float* __restrict__ tgt = target;
    
    for (unsigned int i = idx; i < len; i += stride) {
        tgt[i] = fmaxf(m1[i], m2[i]); // Use restricted pointers
    }
}