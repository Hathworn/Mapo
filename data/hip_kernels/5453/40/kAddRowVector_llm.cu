#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Optimized by calculating the row index directly.
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        unsigned int rowIndex = i / width; // Use width for row index calculation
        tgtMat[i] = mat[i] + vec[rowIndex];
    }
}