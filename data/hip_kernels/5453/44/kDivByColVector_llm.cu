#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDivByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int rowIdx = idx / width; // Calculate row index
    const unsigned int colIdx = idx % width; // Calculate column index

    if (rowIdx < height) {
        tgtMat[idx] = mat[idx] / vec[rowIdx]; // Perform division only if within bounds
    }
}