#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kWhere(float* condition_mat, float* if_mat, float* else_mat, float* target, unsigned int len) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    // Loop unrolling to increase throughput
    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = condition_mat[i] ? if_mat[i] : else_mat[i];
    }
}