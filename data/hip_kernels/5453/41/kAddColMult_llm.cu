#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddColMult(float* mat, float* vec, float* tgtMat, float mult, unsigned int width, unsigned int height) {
    // Calculate thread's linear index within the grid
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of threads across grid
    const unsigned int numThreads = blockDim.x * gridDim.x;
    
    // Use a more precise unsigned long for loop variable to support large matrices
    for (unsigned long i = idx; i < static_cast<unsigned long>(width) * height; i += numThreads) {
        // Precompute index for better efficiency
        unsigned int colIdx = i % height;
        tgtMat[i] = mat[i] + mult * vec[colIdx];
    }
}