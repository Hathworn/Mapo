#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kPowMatrix(float* mat, float* pow, float* target, unsigned int len) {
    // Calculate global thread index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bounds and perform power operation if within range
    if (idx < len) {
        target[idx] = powf(mat[idx], pow[idx]);
    }
}