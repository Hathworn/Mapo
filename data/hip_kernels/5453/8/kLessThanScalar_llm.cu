#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLessThanScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < len) { // Ensure threads do not access out-of-bounds memory
        target[idx] = mat[idx] < val;
    }
}