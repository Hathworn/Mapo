#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultiplyBySigmoidGrad(float* act, float* target, const unsigned int len) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use strided loop with less calculations
    while (idx < len) {
        float act_val = act[idx]; // Cache act value
        target[idx] *= act_val * (1.0f - act_val);
        idx += blockDim.x * gridDim.x;
    }
}