#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetSelectedRows(float* target, float* source, float* indices, int nRowIs, int nCols, int nTargetRows) {
    __shared__ int targetRowIndices[32];
    const int startSourceRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs - startSourceRowI);

    // Load 32 row indices cooperatively
    if (tid < localNRowIs) {
        int idx = int(indices[startSourceRowI + tid]);
        if (idx < 0) idx += nTargetRows;
        targetRowIndices[tid] = (idx < 0 || idx >= nTargetRows) ? -1 : idx;
    }
    __syncthreads();

    // Copy 32 rows optimized by using a separate condition check
    for (int i = 0; i < localNRowIs; i++) {
        const int sourceRowI = startSourceRowI + i;
        const int targetRowI = targetRowIndices[i];
        if (targetRowI != -1) { // Execute only if valid targetRowI
            for (int colI = tid; colI < nCols; colI += 32) {
                target[targetRowI * nCols + colI] = source[sourceRowI * nCols + colI];
            }
        } else if (tid == 0) { // Handle NaN assignment only once per out-of-bounds row
            for (int colI = 0; colI < nCols; colI++) {
                target[targetRowI * nCols + colI] = 1.0 / 0.0 - 1.0 / 0.0;
            }
        }
    }
}