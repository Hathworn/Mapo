#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMultByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate 2D index for matrix elements
    const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < height && col < width) {
        unsigned int index = row * width + col;
        tgtMat[index] = mat[index] * vec[row]; // Multiply with corresponding vector element
    }
}