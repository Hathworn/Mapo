#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize parallelization for better memory access
__global__ void kApplySoftThreshold(float* mat, float alpha, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        float f = mat[idx];
        target[idx] = f > 0 ? max(0.f, f - alpha) : min(0.f, f + alpha); // Use float literals
    }
}