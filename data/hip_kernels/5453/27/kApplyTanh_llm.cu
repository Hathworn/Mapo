#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplyTanh(float* mat, float* target, unsigned int len) {
    // Calculate global thread index.
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Optimize loop to improve memory coalescing
    for (unsigned int i = idx; i < len; i += numThreads) {
        float mat_i = mat[i];
        float exp2x = __expf(2.0f * mat_i);  // Use 2.0f to indicate float operation
        target[i] = 1.0f - 2.0f / (exp2x + 1.0f);  // Use 1.0f for consistency in floats
    }
}