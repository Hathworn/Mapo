#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCorrelate(float* source, float* kernel, float* dest, int width, int height, int kwidth, int kheight) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        float sum = 0;
        int row = i / height;
        int col = i % height;

        // Compute kernel's center offset for indexing
        int half_kwidth = kwidth / 2;
        int half_kheight = kheight / 2;

        // Iterate over kernel
        for (int w = -half_kwidth; w <= half_kwidth; ++w) {
            int current_row = row + w;
            if (current_row < 0 || current_row >= width) continue; // Skip boundary

            for (int h = -half_kheight; h <= half_kheight; ++h) {
                int current_col = col + h;
                if (current_col < 0 || current_col >= height) continue; // Skip boundary

                // Calculate source index
                int src_index = current_row * height + current_col;
                
                // Calculate kernel index
                int kernel_index = (w + half_kwidth) * kheight + (h + half_kheight);
                
                sum += source[src_index] * kernel[kernel_index];
            }
        }
        dest[i] = sum;
    }
}