#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTranspose(float *odata, float *idata, int width, int height) {
    __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE+1]; // Adjusted shared memory size to avoid bank conflicts

    // Calculate global thread positions for reading
    unsigned int xIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;

    // Read input into shared memory
    if ((xIndex < width) && (yIndex < height)) {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    __syncthreads(); // Ensure all reads are done before proceeding

    // Calculate global thread positions for writing
    xIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
    yIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

    // Write output from shared memory to global memory (transposed)
    if ((xIndex < height) && (yIndex < width)) {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}