#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x; // Improved naming for stride

    for (unsigned int i = idx; i < width * height; i += stride) {
        tgtMat[i] = mat[i] + vec[i % height];
    }
}