#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kGreaterThanScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we don't access out-of-bounds memory
    if (idx < len) {
        target[idx] = mat[idx] > val;
    }
}