#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDivide(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use grid-stride loop to efficiently utilize all threads
    for (unsigned int i = idx; i < numEls; i += stride) {
        dest[i] = a[i] / b[i];
    }
}