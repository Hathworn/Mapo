#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultScalar(float* mat, float alpha, float* dest, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if idx is within bounds
    if (idx < len) {
        // Perform computation directly with a single assignment
        dest[idx] = alpha * mat[idx];
    }
}