#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved maximum scalar kernel function
__global__ void kMaximumScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < len) {  // Ensure thread does not exceed array bounds
        target[idx] = fmaxf(mat[idx], val);
    }
}