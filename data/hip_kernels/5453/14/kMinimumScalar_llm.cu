#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMinimumScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Each thread processes multiple elements by striding over the data
    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = fminf(mat[i], val);
    }
}