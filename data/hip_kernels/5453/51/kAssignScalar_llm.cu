#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAssignScalar(float* dest, float alpha, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Unroll loop for better performance
    for (unsigned int i = idx; i < len; i += stride * 4) {
        dest[i] = alpha;
        if (i + stride < len) dest[i + stride] = alpha;
        if (i + 2 * stride < len) dest[i + 2 * stride] = alpha;
        if (i + 3 * stride < len) dest[i + 3 * stride] = alpha;
    }
}