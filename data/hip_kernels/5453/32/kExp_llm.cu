#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kExp(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use stride loop for better memory coalescing 
    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = __expf(mat[i]);
    }
}