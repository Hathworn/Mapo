#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplyAbs(float* mat, float* target, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop through each index stride by total number of threads
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        // Utilize built-in abs function for readability and potential performance gain
        target[i] = fabsf(mat[i]);
    }
}