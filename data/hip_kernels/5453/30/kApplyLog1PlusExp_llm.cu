#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to compute log(1 + exp(x)) for each element in the input array
__global__ void kApplyLog1PlusExp(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    float mat_i;

    for (unsigned int i = idx; i < len; i += numThreads) {
        mat_i = mat[i];
        // Use math function directly to simplify conditional expression
        target[i] = mat_i > 0 ? (__logf(1 + __expf(-mat_i)) + mat_i) : __logf(1 + __expf(mat_i));
    }
}