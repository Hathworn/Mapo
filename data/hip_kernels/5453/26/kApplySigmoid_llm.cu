#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kApplySigmoid(float* mat, float* target, unsigned int len) {
    // Calculate global index for current thread
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop for better parallelism
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = 1.0f / (1.0f + __expf(-mat[i]));  // Use float literals for precision
    }
}