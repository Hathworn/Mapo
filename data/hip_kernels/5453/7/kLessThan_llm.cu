#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLessThan(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure idx does not exceed length of arrays
    if (idx < len) {
        target[idx] = mat1[idx] < mat2[idx];
    }
}