#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSubtract(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate the thread's absolute index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure each thread operates within bounds
    if (idx < numEls) {
        dest[idx] = a[idx] - b[idx];
    }
}