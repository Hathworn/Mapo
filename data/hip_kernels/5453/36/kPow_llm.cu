#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kPow(float* __restrict__ mat, float pow, float* __restrict__ target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;  // Use 'stride' instead of 'numThreads'

    // Use shared memory for more efficient memory access
    extern __shared__ float sharedMat[];
    sharedMat[threadIdx.x] = mat[idx];
    __syncthreads();

    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = powf(sharedMat[threadIdx.x], pow);  // Compute using shared memory
    }
}