#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMult(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Check bounds to avoid unnecessary calculations
    if (idx < numEls) {
        dest[idx] = a[idx] * b[idx];
    }
}