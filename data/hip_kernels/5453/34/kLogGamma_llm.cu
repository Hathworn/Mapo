#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogGamma(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use strided loop with shared memory to minimize global memory access
    for (unsigned int i = idx; i < len; i += gridDim.x * blockDim.x) {
        target[i] = lgammaf(mat[i]);
    }
}