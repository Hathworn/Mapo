#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate unique thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of threads in the grid
    const unsigned int stride = blockDim.x * gridDim.x;
    unsigned int colIndex, rowIndex;

    // Loop over elements with strided access pattern
    for (unsigned int i = idx; i < width * height; i += stride) {
        rowIndex = i % height;  // Calculate row index
        tgtMat[i] = mat[i] * vec[rowIndex];  // Multiply by corresponding vector element
    }
}