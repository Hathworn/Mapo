#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kGamma(float* __restrict__ mat, float* __restrict__ target, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure that no extra threads do unnecessary work
    if (idx < len) {
        target[idx] = tgammaf(mat[idx]);
    }
}