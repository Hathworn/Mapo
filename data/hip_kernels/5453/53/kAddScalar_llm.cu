#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddScalar(float* a, float alpha, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure threads do not exceed array bounds
    if (idx < numEls) {
        dest[idx] = a[idx] + alpha;
    }
}