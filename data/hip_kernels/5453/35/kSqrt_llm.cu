#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSqrt(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x; // Simplify with stride instead of numThreads

    // Use 'stride' instead of incrementing by numThreads
    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = sqrtf(mat[i]); // Use 'sqrtf' for better performance with 'float'
    }
}