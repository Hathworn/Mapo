#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kReciprocal(float* mat, float* target, unsigned int len) {
    // Calculate linear thread index across the grid
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use each thread to handle a single element if within length
    if (idx < len) {
        target[idx] = 1.f / mat[idx];
    }
}