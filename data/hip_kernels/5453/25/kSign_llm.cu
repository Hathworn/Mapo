#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSign(float* mat, float* target, unsigned int len) {
// Calculate global index
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

// Ensure thread index is within bounds
if (idx < len) {
    target[idx] = mat[idx] ? copysignf(1.f, mat[idx]) : 0.f;
}
}