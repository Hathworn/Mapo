#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate row and column indices
    unsigned int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (col < width) {  // Ensure column index is within bounds
        for (unsigned int row = 0; row < height; ++row) {  // Iterate over each row
            unsigned int idx = row * width + col;
            tgtMat[idx] = mat[idx] / vec[row];
        }
    }
}