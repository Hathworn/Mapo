#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global thread ID for 2D grid
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Conditional check for valid index in bounds
    if (i < h && j < w)
    {
        // Calculate linear index
        int pos = i * w + j;
        // Set value in image
        image[pos] = value;
    }
}