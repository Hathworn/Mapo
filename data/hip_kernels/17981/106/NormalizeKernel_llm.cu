#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate the global thread indices
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds to ensure threads only operate on valid elements
    if (i < h && j < w)
    {
        const int pos = i * s + j;
        float scale = normalization_factor[pos];
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);
        image[pos] *= invScale;
    }
}