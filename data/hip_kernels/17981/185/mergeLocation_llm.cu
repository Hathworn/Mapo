#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for coalesced memory access pattern if applicable
    if (ptidx < npoints)
    {
        // Load loc_ data into registers to reduce global memory access
        short2 loc = loc_[ptidx];

        // Direct computation without storing intermediate results
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}