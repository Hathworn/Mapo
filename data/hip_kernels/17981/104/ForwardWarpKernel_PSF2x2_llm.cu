#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // pixel containing bottom left corner
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    // target pixel integer coords
    int tx = (int) px;
    int ty = (int) py;
    float value = src[image_row_offset + j];

    // fill pixel containing bottom right corner
    if ((tx < w) && (tx >= 0) && (ty < h) && (ty >= 0))
    {
        float weight = dx * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // fill pixel containing bottom left corner
    tx -= 1;
    if ((tx < w) && (tx >= 0) && (ty < h) && (ty >= 0))
    {
        float weight = (1.0f - dx) * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // fill pixel containing upper left corner
    ty -= 1;
    if ((tx < w) && (tx >= 0) && (ty < h) && (ty >= 0))
    {
        float weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // fill pixel containing upper right corner
    tx += 1;
    if ((tx < w) && (tx >= 0) && (ty < h) && (ty >= 0))
    {
        float weight = dx * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}