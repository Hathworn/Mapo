#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Use local variable to store thread position for performance
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check boundary within the main computation block
    if (pos < len) {
        d_res[pos] = d_src[pos] * scale;
    }
}