#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void add(int n, float *a, float *b, float *sum)
{
    // Use shared memory for partial results to optimize memory access
    extern __shared__ float shared[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < n)
    {
        shared[threadIdx.x] = a[i] + 2 * b[i];
        __syncthreads(); // Ensure all threads have completed their work 

        // Write the final result back to global memory
        sum[i] = shared[threadIdx.x];
    }
}