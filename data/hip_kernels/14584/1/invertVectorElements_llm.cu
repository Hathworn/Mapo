#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void invertVectorElements(float* vector, int n)
{
    // Use blockIdx and blockDim to handle larger vectors
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Avoid division by zero
        float val = vector[i];
        if (val != 0.0f)
        {
            vector[i] = 1.0f / val;
        }
    }
}