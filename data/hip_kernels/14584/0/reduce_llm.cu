#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void reduce(float *g_idata, float *g_odata, unsigned int n) {
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int gridSize = blockDim.x * 2 * gridDim.x;

    float mySum = 0;

    // Loop unrolled to improve memory coalescing
    if (i < n) mySum = g_idata[i];
    if (i + blockDim.x < n) mySum += g_idata[i + blockDim.x];
    i += gridSize;

    // Store the local sum in shared memory
    sdata[tid] = mySum;
    __syncthreads();

    // Reduce in shared memory using coalesced access
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Unrolling the last warp
    if (tid < 32) {
        volatile float *smem = sdata;
        smem[tid] += smem[tid + 32];
        smem[tid] += smem[tid + 16];
        smem[tid] += smem[tid + 8];
        smem[tid] += smem[tid + 4];
        smem[tid] += smem[tid + 2];
        smem[tid] += smem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}