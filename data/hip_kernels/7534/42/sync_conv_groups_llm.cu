#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function to handle synchronization
__global__ void sync_conv_groups() {
    __syncthreads(); // Ensure that threads within a block are synchronized
}