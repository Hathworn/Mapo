#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gMaxPoolingForward(float* out, int outRows, int outCols, float* in, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= outRows * outCols)
        return;

    int rowId = tid / outCols; // Corrected calculation to match outCols
    int colId = tid % outCols; // Corrected calculation to match outCols

    float* b = in + (rowId * inCols) + (colId * width);
    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;

    if (colId == outCols - 1) { // Corrected check to match outCols
        width = lastWidth;
    }

    float currentMax = -FLT_MAX; // Initialize with negative infinity for robustness
    for (int i = 0; i < width; ++i) { // Start from 0 for correct comparison
        float val = b[i] * localMask[i]; // Calculate once per iteration
        if (val > currentMax) {
            currentMax = val;
        }
    }

    out[rowId * outCols + colId] = currentMax; // Corrected indexing for correct placement
}