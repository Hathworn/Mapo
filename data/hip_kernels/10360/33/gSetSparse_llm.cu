#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gSetSparse(float* out, const size_t* indices, const float* values, int length) {
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride loop for improved memory access pattern
    for(; index < length; index += blockDim.x * gridDim.x) {
        out[indices[index]] = values[index];
    }
}