#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gGetValueByKey(float* d_in, float* d_out, int* indeces, int n) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    // Check boundary condition with warp size step
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        int index = indeces[i];
        d_out[i] = d_in[index];
    }
}