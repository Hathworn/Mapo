#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gScatterAdd(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    // Calculate the global thread index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Use a shared memory buffer to coalesce the access to denseData
    __shared__ float cache[256]; // Assuming optimal block size of 256

    if (idx < sparseSize) {
        int denseIndex = sparseIndices[idx] + offset;

        // Check bounds and perform scatter add
        if(denseIndex >= 0 && denseIndex < denseSize) {
            int cacheIndex = threadIdx.x;
            cache[cacheIndex] = sparseData[idx];

            // Use atomic add to ensure no race conditions occur
            atomicAdd(&denseData[denseIndex], cache[cacheIndex]);
        }
    }
}