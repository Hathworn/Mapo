#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gScatterUpdate(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    // Calculate flattened global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use branchless logic for bounds checking
    if (idx < sparseSize) {
        int sparseIndex = sparseIndices[idx] + offset;
        if (sparseIndex >= 0 && sparseIndex < denseSize) {
            // Perform scatter update
            denseData[sparseIndex] = sparseData[idx];
        }
    }
}