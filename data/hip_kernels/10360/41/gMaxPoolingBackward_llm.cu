#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gMaxPoolingBackward(float* adj, int adjRows, int adjCols, float* in, float* adjIn, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= adjRows * adjCols)
        return;

    // Calculate row and column index for simplicity
    int rowId = tid / adjRows;
    int colId = tid % adjRows;

    int currentWidth = (colId == adjRows - 1) ? lastWidth : width;  // Use lastWidth if at the last column

    float* b = in + (rowId * inCols) + (colId * width);
    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;

    // Initialize currentMaxIdx for finding the maximum value
    size_t currentMaxIdx = 0;
    float maxVal = b[0] * localMask[0];
    for (int i = 1; i < currentWidth; ++i) {
        float val = b[i] * localMask[i];
        if (val > maxVal) {
            maxVal = val;
            currentMaxIdx = i;
        }
    }

    // Use atomic addition to prevent race conditions
    atomicAdd(&adjIn[(rowId * inCols) + (colId * width) + currentMaxIdx],
              adj[rowId + (colId * adjCols)]);
}