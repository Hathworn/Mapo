#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel with coalesced memory access
__global__ void gGather(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure idx is within bounds
    if(idx < sparseSize) {
        int sparseIdx = sparseIndices[idx] + offset;

        // Check boundaries for valid memory access
        if(sparseIdx >= 0 && sparseIdx < denseSize) {
            sparseData[idx] = denseData[sparseIdx];
        }
    }
}