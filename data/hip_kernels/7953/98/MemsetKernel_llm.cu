#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Return early if thread is out of bounds
    if (i >= h || j >= w) return;
    
    const int pos = i * w + j;

    // Use shared memory for coalesced writes if necessary
    __shared__ float shared_value;
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        shared_value = value;
    }
    __syncthreads();

    image[pos] = shared_value;
}