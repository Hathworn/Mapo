#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];

        // Use local variables to reduce memory access
        float scaled_x = loc.x * scale;
        float scaled_y = loc.y * scale;

        x[ptidx] = scaled_x;
        y[ptidx] = scaled_y;
    }
}