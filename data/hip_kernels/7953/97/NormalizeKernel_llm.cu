#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    if (i >= h || j >= w) return;

    const int pos = i * s + j;

    float scale = normalization_factor[pos];

    // Precompute the inverse scale efficiently
    float invScale = __fdividef(1.0f, scale);

    image[pos] *= invScale;
}