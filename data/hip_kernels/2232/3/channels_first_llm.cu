#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void channels_first(float* input, float* rinput, int channels, int height, int width, int pad_size)
{
    // Use shared memory for better memory access pattern if needed
    __shared__ float shared_mem[THREADS_PER_BLOCK];

    // Calculate the global position
    int n = blockIdx.x;
    int y = blockIdx.y;
    int x = blockIdx.z;
    int ch_off = threadIdx.x;

    // Compute dimensions
    int dimcyx = channels * height * width;
    int dimyx = height * width;
    int p_dimx = (width + 2 * pad_size);
    int p_dimy = (height + 2 * pad_size);
    int p_dimyxc = channels * p_dimy * p_dimx;
    int p_dimxc = p_dimx * channels;

    // Calculate input and rinput offsets
    int input_offset = n * dimcyx + y * width + x;
    int rinput_offset = n * p_dimyxc + (y + pad_size) * p_dimxc + (x + pad_size) * channels;

    // Unroll loop if possible for better performance
    for (int c = ch_off; c < channels; c += THREADS_PER_BLOCK) {
        // Use shared memory if accessed multiple times
        shared_mem[ch_off] = input[input_offset + c * dimyx];
        __syncthreads(); // Synchronize threads if necessary

        rinput[rinput_offset + c] = shared_mem[ch_off];
    }
}