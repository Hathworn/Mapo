#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Correlation_forward( float *output, int nOutputChannels, int outputHeight, int outputWidth, float *rInput1, int nInputChannels, int inputHeight, int inputWidth, float *rInput2, int pad_size, int kernel_size, int max_displacement, int stride1, int stride2)
{
    // n (batch size), c (num of channels), y (height), x (width)

    int pInputWidth = inputWidth + 2 * pad_size;
    int pInputHeight = inputHeight + 2 * pad_size;

    int kernel_rad = (kernel_size - 1) / 2;
    int displacement_rad = max_displacement / stride2;
    int displacement_size = 2 * displacement_rad + 1;

    int n  = blockIdx.x;
    int y1 = blockIdx.y * stride1 + max_displacement + kernel_rad;
    int x1 = blockIdx.z * stride1 + max_displacement + kernel_rad;
    int c = threadIdx.x;

    int pdimyxc = pInputHeight * pInputWidth * nInputChannels;
    int pdimxc = pInputWidth * nInputChannels;
    int pdimc = nInputChannels;

    int tdimcyx = nOutputChannels * outputHeight * outputWidth;
    int tdimyx = outputHeight * outputWidth;
    int tdimx = outputWidth;

    float nelems = kernel_size * kernel_size * pdimc;

    __shared__ float prod_sum[THREADS_PER_BLOCK];

    // Element-wise product along channel axis. Initialize prod_sum
    prod_sum[c] = 0;

    // Loop unrolling for better performance
    for (int tj = -displacement_rad; tj <= displacement_rad; ++tj) {
        for (int ti = -displacement_rad; ti <= displacement_rad; ++ti) {
            int x2 = x1 + ti * stride2;
            int y2 = y1 + tj * stride2;

            // Precompute index start to reduce redundant calculation
            int idx_start1 = n * pdimyxc + y1 * pdimxc + x1 * pdimc;
            int idx_start2 = n * pdimyxc + y2 * pdimxc + x2 * pdimc;

            for (int j = -kernel_rad; j <= kernel_rad; ++j) {
                int idx1 = idx_start1 + j * pdimxc;
                int idx2 = idx_start2 + j * pdimxc;

                for (int i = -kernel_rad; i <= kernel_rad; ++i) {
                    for (int ch = c; ch < pdimc; ch += THREADS_PER_BLOCK) {
                        prod_sum[c] += rInput1[idx1 + i * pdimc + ch] * rInput2[idx2 + i * pdimc + ch];
                    }
                }
            }

            // Sum reduction using parallel threads
            __syncthreads();
            if (c == 0) {
                float reduce_sum = 0;
                for (int index = 0; index < THREADS_PER_BLOCK; ++index) {
                    reduce_sum += prod_sum[index];
                }
                int tc = (tj + displacement_rad) * displacement_size + (ti + displacement_rad);
                const int tindx = n * tdimcyx + tc * tdimyx + blockIdx.y * tdimx + blockIdx.z;
                output[tindx] = reduce_sum / nelems;
            }
        }
    }
}