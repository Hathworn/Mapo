#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NmDistanceGradKernel(int b, int n, const float *xyz1, int m, const float *xyz2, const float *grad_dist1, const int *idx1, float *grad_xyz1, float *grad_xyz2) {
    // Optimize index calculations using reusable variables
    int batchIdx = blockIdx.x * gridDim.y + blockIdx.y;
    if (batchIdx >= b) return;

    int pointIdx = threadIdx.x + blockIdx.y * blockDim.x;
    while (pointIdx < n) {
        int idx1_offset = batchIdx * n + pointIdx;
        int idx2 = idx1[idx1_offset];
        
        float x1 = xyz1[idx1_offset * 2];
        float y1 = xyz1[idx1_offset * 2 + 1];
        float x2 = xyz2[(batchIdx * m + idx2) * 2];
        float y2 = xyz2[(batchIdx * m + idx2) * 2 + 1];
        
        float g = grad_dist1[idx1_offset] * 2;

        // Reduce global memory access and improve data locality
        atomicAdd(&grad_xyz1[idx1_offset * 2], g * (x1 - x2));
        atomicAdd(&grad_xyz1[idx1_offset * 2 + 1], g * (y1 - y2));
        atomicAdd(&grad_xyz2[(batchIdx * m + idx2) * 2], -(g * (x1 - x2)));
        atomicAdd(&grad_xyz2[(batchIdx * m + idx2) * 2 + 1], -(g * (y1 - y2)));

        pointIdx += blockDim.x * gridDim.y;
    }
}