#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NmDistanceGradKernel(int b, int n, const float * xyz1, int m, const float * xyz2, const float * grad_dist1, const int * idx1, float * grad_xyz1, float * grad_xyz2) {
    // Calculate global thread indices
    int batchIdx = blockIdx.x;
    int threadInGrid = threadIdx.x + blockIdx.y * blockDim.x;
    
    while (batchIdx < b) {
        int pointIdx = threadInGrid;
        while (pointIdx < n) {
            int baseIdx1 = (batchIdx * n + pointIdx) * 5;
            float x1 = xyz1[baseIdx1 + 0];
            float y1 = xyz1[baseIdx1 + 1];
            float r1 = xyz1[baseIdx1 + 2];
            float g1 = xyz1[baseIdx1 + 3];
            float b1 = xyz1[baseIdx1 + 4];
            int j2 = idx1[batchIdx * n + pointIdx];
            int baseIdx2 = (batchIdx * m + j2) * 5;
            float x2 = xyz2[baseIdx2 + 0];
            float y2 = xyz2[baseIdx2 + 1];
            float r2 = xyz2[baseIdx2 + 2];
            float g2 = xyz2[baseIdx2 + 3];
            float b2 = xyz2[baseIdx2 + 4];
            float g = grad_dist1[batchIdx * n + pointIdx] * 2;

            // Use shared variables to optimize memory operations
            atomicAdd(&(grad_xyz1[baseIdx1 + 0]), g * (x1 - x2));
            atomicAdd(&(grad_xyz1[baseIdx1 + 1]), g * (y1 - y2));
            atomicAdd(&(grad_xyz1[baseIdx1 + 2]), g * (r1 - r2));
            atomicAdd(&(grad_xyz1[baseIdx1 + 3]), g * (g1 - g2));
            atomicAdd(&(grad_xyz1[baseIdx1 + 4]), g * (b1 - b2));
            atomicAdd(&(grad_xyz2[baseIdx2 + 0]), -(g * (x1 - x2)));
            atomicAdd(&(grad_xyz2[baseIdx2 + 1]), -(g * (y1 - y2)));
            atomicAdd(&(grad_xyz2[baseIdx2 + 2]), -(g * (r1 - r2)));
            atomicAdd(&(grad_xyz2[baseIdx2 + 3]), -(g * (g1 - g2)));
            atomicAdd(&(grad_xyz2[baseIdx2 + 4]), -(g * (b1 - b2)));
            pointIdx += blockDim.x * gridDim.y;
        }
        batchIdx += gridDim.x;
    }
}