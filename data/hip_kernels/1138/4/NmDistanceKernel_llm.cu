#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NmDistanceKernel(int b, int n, const float * xyz, int m, const float * xyz2, float * result, int * result_i) {
    const int batch = 512;
    __shared__ float buf[batch * 2];

    for (int i = blockIdx.x; i < b; i += gridDim.x) {
        for (int k2 = 0; k2 < m; k2 += batch) {
            int end_k = min(m, k2 + batch) - k2;

            // Load xyz2 data into shared memory
            for (int j = threadIdx.x; j < end_k * 2; j += blockDim.x) {
                buf[j] = xyz2[(i * m + k2) * 2 + j];
            }
            __syncthreads();

            for (int j = threadIdx.x + blockIdx.y * blockDim.x; j < n; j += blockDim.x * gridDim.y) {
                float x1 = xyz[(i * n + j) * 2 + 0];
                float y1 = xyz[(i * n + j) * 2 + 1];
                float best = FLT_MAX;  // Initialize with maximum float value
                int best_i = 0;

                // Unroll loop to improve performance
                int k;
                for (k = 0; k <= end_k - 4; k += 4) {
                    #pragma unroll
                    for (int offset = 0; offset < 4; ++offset) {
                        float x2 = buf[(k + offset) * 2 + 0] - x1;
                        float y2 = buf[(k + offset) * 2 + 1] - y1;
                        float d = x2 * x2 + y2 * y2;
                        if (d < best) {
                            best = d;
                            best_i = k + offset + k2;
                        }
                    }
                }

                // Handle remaining elements
                for (; k < end_k; ++k) {
                    float x2 = buf[k * 2 + 0] - x1;
                    float y2 = buf[k * 2 + 1] - y1;
                    float d = x2 * x2 + y2 * y2;
                    if (d < best) {
                        best = d;
                        best_i = k + k2;
                    }
                }

                // Update the result if necessary
                int result_idx = i * n + j;
                if (k2 == 0 || result[result_idx] > best) {
                    result[result_idx] = best;
                    result_i[result_idx] = best_i;
                }
            }
            __syncthreads();
        }
    }
}