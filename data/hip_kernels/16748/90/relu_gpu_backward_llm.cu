#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    // Calculate thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread ID is within bounds and directly set ingrad based on condition
    if (tid < N) {
        ingrad[tid] = (indata[tid] > 0) * outgrad[tid];
    }
}