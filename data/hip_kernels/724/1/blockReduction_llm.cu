#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blockReduction(double* dN_pTdpTdphidy_d, int final_spectrum_size, int blocks_ker1)
{
    long idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < final_spectrum_size)
    {
        if (blocks_ker1 == 1) return; // Early exit if only one block

        double sum = dN_pTdpTdphidy_d[idx];
        for (int i = 1; i < blocks_ker1; i++)
        {
            sum += dN_pTdpTdphidy_d[idx + i * final_spectrum_size];
        }
        dN_pTdpTdphidy_d[idx] = sum;

        if (isnan(sum)) printf("found dN_pTdpTdphidy_d nan \n"); // Check NaN after reduction
    }
}