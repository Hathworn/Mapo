#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifndef _ICP_KERNEL_H_
#define _ICP_KERNEL_H_

#define TILE_WIDTH 256

#endif // #ifndef _ICP_KERNEL_H_

__global__ void CalculateTotalError(double * distance_d, int size_data)
{
    __shared__ double error_s[2 * TILE_WIDTH];
    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockDim.x * blockIdx.x;
    
    // Load data into shared memory with boundary checks
    error_s[t] = (start + t < size_data) ? distance_d[start + t] : 0.0;
    error_s[blockDim.x + t] = (start + blockDim.x + t < size_data) ? distance_d[start + blockDim.x + t] : 0.0;
    
    // Optimized reduction loop
    for (unsigned int stride = blockDim.x; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (t < stride)
        {
            error_s[t] += error_s[t + stride];
        }
    }
    
    // Write the result of this block to global memory
    if (t == 0)
    {
        distance_d[blockIdx.x] = error_s[0];
    }
}