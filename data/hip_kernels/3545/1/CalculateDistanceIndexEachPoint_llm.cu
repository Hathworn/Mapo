#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifndef _ICP_KERNEL_H_
#define _ICP_KERNEL_H_

#define TILE_WIDTH 256

__global__ void CalculateDistanceIndexEachPoint(double point_x, double point_y, double point_z, double *data_x_d, double *data_y_d, double *data_z_d, int *bin_index_d, double *distance_d, int size_data) 
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Check if index is within data bounds
    if (index < size_data) {
        double dx = data_x_d[index] - point_x;  // Calculate x distance
        double dy = data_y_d[index] - point_y;  // Calculate y distance
        double dz = data_z_d[index] - point_z;  // Calculate z distance

        // Store computed distance
        distance_d[index] = sqrt(dx * dx + dy * dy + dz * dz); 

        // Assign index
        bin_index_d[index] = index; 
    }
}

#endif // #ifndef _ICP_KERNEL_H_