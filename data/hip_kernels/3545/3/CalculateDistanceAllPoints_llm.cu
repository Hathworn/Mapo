#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifndef _ICP_KERNEL_H_
#define _ICP_KERNEL_H_

#define TILE_WIDTH 256

#endif // #ifndef _ICP_KERNEL_H_

__global__ void CalculateDistanceAllPoints(double * data_x_d, double * data_y_d, double * data_z_d, double * transformed_data_x_d, double * transformed_data_y_d, double * transformed_data_z_d, int * index_d, double * distance_d, int size_data)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if(i < size_data)
    {
        int index = index_d[i];

        // Calculate squared distances first to avoid repeated computation
        double dx = data_x_d[index] - transformed_data_x_d[i];
        double dy = data_y_d[index] - transformed_data_y_d[i];
        double dz = data_z_d[index] - transformed_data_z_d[i];

        // Use constant memory when possible and minimize expensive sqrt operation
        distance_d[i] = sqrt(dx * dx + dy * dy + dz * dz);
    }
}