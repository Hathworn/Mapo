#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifndef _ICP_KERNEL_H_
#define _ICP_KERNEL_H_

#define TILE_WIDTH 256

#endif // #ifndef _ICP_KERNEL_H_

__global__ void CalculateBestIndex(double * distance_d, int * bin_index_d, int size_data)
{
    // Use shared memory to store distances and indices
    __shared__ double distance_s[2*TILE_WIDTH];
    __shared__ unsigned int bin_smallest_index[2*TILE_WIDTH];

    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockDim.x * blockIdx.x;

    if(start + t < size_data)
    {
        distance_s[t] = distance_d[start + t];
        bin_smallest_index[t] = bin_index_d[start + t];
    }
    else
    {
        distance_s[t] = 65535;
        bin_smallest_index[t] = 0;
    }
    
    if(start + blockDim.x + t < size_data)
    {
        distance_s[blockDim.x + t] = distance_d[start + blockDim.x + t];
        bin_smallest_index[blockDim.x + t] = bin_index_d[start + blockDim.x + t];
    }
    else
    {
        distance_s[blockDim.x + t] = 65535;
        bin_smallest_index[blockDim.x + t] = 0;
    }

    // Perform parallel reduction in shared memory
    for(unsigned int stride = blockDim.x; stride >= 1; stride >>= 1)
    {
        __syncthreads();
        if(t < stride)
        {
            if(distance_s[t] > distance_s[stride + t])
            {
                bin_smallest_index[t] = bin_smallest_index[stride + t];
                distance_s[t] = distance_s[stride + t];
            }
        }
    }

    // Write results for this block to global memory
    if(t == 0)
    {
        distance_d[blockIdx.x] = distance_s[0];
        bin_index_d[blockIdx.x] = bin_smallest_index[0];
    }
}