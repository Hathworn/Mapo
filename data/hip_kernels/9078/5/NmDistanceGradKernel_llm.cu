#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NmDistanceGradKernel(int b, int n, const float * xyz1, int m, const float * xyz2, const float * grad_dist1, const int * idx1, float * grad_xyz1, float * grad_xyz2) {
    int i = blockIdx.x;  // Each block works on its own batch
    int j_start = threadIdx.x + blockIdx.y * blockDim.x; // Calculate starting index for this thread

    for (int j = j_start; j < n; j += blockDim.x * gridDim.y) {
        float x1 = xyz1[(i * n + j) * 3 + 0];
        float y1 = xyz1[(i * n + j) * 3 + 1];
        float z1 = xyz1[(i * n + j) * 3 + 2];
        
        int j2 = idx1[i * n + j];
        
        float x2 = xyz2[(i * m + j2) * 3 + 0];
        float y2 = xyz2[(i * m + j2) * 3 + 1];
        float z2 = xyz2[(i * m + j2) * 3 + 2];
        
        float g = grad_dist1[i * n + j] * 2;
        
        // Precompute g * (x1 - x2), g * (y1 - y2), g * (z1 - z2) to avoid redundancy
        float diff_x = g * (x1 - x2);
        float diff_y = g * (y1 - y2);
        float diff_z = g * (z1 - z2);
        
        atomicAdd(&(grad_xyz1[(i * n + j) * 3 + 0]), diff_x);
        atomicAdd(&(grad_xyz1[(i * n + j) * 3 + 1]), diff_y);
        atomicAdd(&(grad_xyz1[(i * n + j) * 3 + 2]), diff_z);
        
        atomicAdd(&(grad_xyz2[(i * m + j2) * 3 + 0]), -diff_x);
        atomicAdd(&(grad_xyz2[(i * m + j2) * 3 + 1]), -diff_y);
        atomicAdd(&(grad_xyz2[(i * m + j2) * 3 + 2]), -diff_z);
    }
}