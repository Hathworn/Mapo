#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NmDistanceKernel(int b, int n, const float *xyz, int m, const float *xyz2, float *result, int *result_i) {
    const int batch = 512;
    __shared__ float buf[batch * 3];

    for (int i = blockIdx.x; i < b; i += gridDim.x) {
        for (int k2 = 0; k2 < m; k2 += batch) {
            int end_k = min(m, k2 + batch) - k2;

            // Load batch of xyz2 into shared memory
            for (int j = threadIdx.x; j < end_k * 3; j += blockDim.x) {
                buf[j] = xyz2[(i * m + k2) * 3 + j];
            }
            __syncthreads();

            // Parallelize over the `n` dimensions
            for (int j = threadIdx.x + blockIdx.y * blockDim.x; j < n; j += blockDim.x * gridDim.y) {
                float x1 = xyz[(i * n + j) * 3 + 0];
                float y1 = xyz[(i * n + j) * 3 + 1];
                float z1 = xyz[(i * n + j) * 3 + 2];
                int best_i = 0;
                float best = FLT_MAX;

                // Unroll 4 times
                for (int k = 0; k < end_k; k += 4) {
                    if (k + 0 < end_k) { // Check boundary for unrolling
                        float x2 = buf[(k + 0) * 3 + 0] - x1;
                        float y2 = buf[(k + 0) * 3 + 1] - y1;
                        float z2 = buf[(k + 0) * 3 + 2] - z1;
                        float d = x2 * x2 + y2 * y2 + z2 * z2;
                        if (d < best) {
                            best = d;
                            best_i = k + k2;
                        }
                    }
                    if (k + 1 < end_k) {
                        float x2 = buf[(k + 1) * 3 + 0] - x1;
                        float y2 = buf[(k + 1) * 3 + 1] - y1;
                        float z2 = buf[(k + 1) * 3 + 2] - z1;
                        float d = x2 * x2 + y2 * y2 + z2 * z2;
                        if (d < best) {
                            best = d;
                            best_i = k + k2 + 1;
                        }
                    }
                    if (k + 2 < end_k) {
                        float x2 = buf[(k + 2) * 3 + 0] - x1;
                        float y2 = buf[(k + 2) * 3 + 1] - y1;
                        float z2 = buf[(k + 2) * 3 + 2] - z1;
                        float d = x2 * x2 + y2 * y2 + z2 * z2;
                        if (d < best) {
                            best = d;
                            best_i = k + k2 + 2;
                        }
                    }
                    if (k + 3 < end_k) {
                        float x2 = buf[(k + 3) * 3 + 0] - x1;
                        float y2 = buf[(k + 3) * 3 + 1] - y1;
                        float z2 = buf[(k + 3) * 3 + 2] - z1;
                        float d = x2 * x2 + y2 * y2 + z2 * z2;
                        if (d < best) {
                            best = d;
                            best_i = k + k2 + 3;
                        }
                    }
                }

                if (k2 == 0 || result[i * n + j] > best) {
                    result[i * n + j] = best;
                    result_i[i * n + j] = best_i;
                }
            }
            __syncthreads();
        }
    }
}