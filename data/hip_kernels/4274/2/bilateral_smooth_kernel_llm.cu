#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 1e-4

// Optimize global memory access using shared memory
__global__ void bilateral_smooth_kernel(float *affine_model, float *filtered_affine_model, float *guide, int h, int w, int kernel_radius, float sigma1, float sigma2) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;
    
    // Declare shared memory for guide
    __shared__ float shared_guide[TB * 3];
    
    if (id < size) {
        int x = id % w;
        int y = id / w;

        double sum_affine[12] = {};
        double sum_weight = 0;
        
        // Load guide data into shared memory for current block
        for (int i = 0; i < 3; i++) {
            shared_guide[threadIdx.x * 3 + i] = guide[id + i * size];
        }
        __syncthreads();
        
        for (int dx = -kernel_radius; dx <= kernel_radius; dx++) {
            for (int dy = -kernel_radius; dy <= kernel_radius; dy++) {
                int yy = y + dy, xx = x + dx;
                int id2 = yy * w + xx;
                if (0 <= xx && xx < w && 0 <= yy && yy < h) {
                    // Calculate color differences using shared memory
                    float color_diff1 = shared_guide[threadIdx.x * 3] - shared_guide[(yy*w + xx) % TB * 3];
                    float color_diff2 = shared_guide[threadIdx.x * 3 + 1] - shared_guide[(yy*w + xx) % TB * 3 + 1];
                    float color_diff3 = shared_guide[threadIdx.x * 3 + 2] - shared_guide[(yy*w + xx) % TB * 3 + 2];
                    float color_diff_sqr = (color_diff1 * color_diff1 + color_diff2 * color_diff2 + color_diff3 * color_diff3) / 3;

                    float v1 = exp(-(dx * dx + dy * dy) / (2 * sigma1 * sigma1));
                    float v2 = exp(-(color_diff_sqr) / (2 * sigma2 * sigma2));
                    float weight = v1 * v2;

                    // Use cache-friendly memory access
                    for (int i = 0; i < 3; i++) {
                        for (int j = 0; j < 4; j++) {
                            int affine_id = i * 4 + j;
                            sum_affine[affine_id] += weight * affine_model[id2 * 12 + affine_id];
                        }
                    }
                    sum_weight += weight;
                }
            }
        }

        // Update filtered affine model
        for (int i = 0; i < 3; i++) {
            for (int j = 0; j < 4; j++) {
                int affine_id = i * 4 + j;
                filtered_affine_model[id * 12 + affine_id] = sum_affine[affine_id] / sum_weight;
            }
        }
    }
}