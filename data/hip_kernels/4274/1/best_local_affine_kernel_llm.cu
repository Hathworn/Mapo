#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 1e-4

__device__ bool InverseMat4x4(double m_in[4][4], double inv_out[4][4]) {
    double m[16], inv[16];
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            m[i * 4 + j] = m_in[i][j];
        }
    }

    inv[0] = m[5]  * m[10] * m[15] - m[5]  * m[11] * m[14] - m[9]  * m[6] * m[15] + m[9]  * m[7] * m[14] + m[13] * m[6] * m[11] - m[13] * m[7] * m[10];
    inv[4] = -m[4]  * m[10] * m[15] + m[4]  * m[11] * m[14] + m[8]  * m[6] * m[15] - m[8]  * m[7] * m[14] - m[12] * m[6] * m[11] + m[12] * m[7] * m[10];
    inv[8] = m[4]  * m[9] * m[15] - m[4]  * m[11] * m[13] - m[8]  * m[5] * m[15] + m[8]  * m[7] * m[13] + m[12] * m[5] * m[11] - m[12] * m[7] * m[9];
    inv[12] = -m[4]  * m[9] * m[14] + m[4]  * m[10] * m[13] + m[8]  * m[5] * m[14] - m[8]  * m[6] * m[13] - m[12] * m[5] * m[10] + m[12] * m[6] * m[9];
    inv[1] = -m[1]  * m[10] * m[15] + m[1]  * m[11] * m[14] + m[9]  * m[2] * m[15] - m[9]  * m[3] * m[14] - m[13] * m[2] * m[11] + m[13] * m[3] * m[10];
    inv[5] = m[0]  * m[10] * m[15] - m[0]  * m[11] * m[14] - m[8]  * m[2] * m[15] + m[8]  * m[3] * m[14] + m[12] * m[2] * m[11] - m[12] * m[3] * m[10];
    inv[9] = -m[0]  * m[9] * m[15] + m[0]  * m[11] * m[13] + m[8]  * m[1] * m[15] - m[8]  * m[3] * m[13] - m[12] * m[1] * m[11] + m[12] * m[3] * m[9];
    inv[13] = m[0]  * m[9] * m[14] - m[0]  * m[10] * m[13] - m[8]  * m[1] * m[14] + m[8]  * m[2] * m[13] + m[12] * m[1] * m[10] - m[12] * m[2] * m[9];
    inv[2] = m[1]  * m[6] * m[15] - m[1]  * m[7] * m[14] - m[5]  * m[2] * m[15] + m[5]  * m[3] * m[14] + m[13] * m[2] * m[7] - m[13] * m[3] * m[6];
    inv[6] = -m[0]  * m[6] * m[15] + m[0]  * m[7] * m[14] + m[4]  * m[2] * m[15] - m[4]  * m[3] * m[14] - m[12] * m[2] * m[7] + m[12] * m[3] * m[6];
    inv[10] = m[0]  * m[5] * m[15] - m[0]  * m[7] * m[13] - m[4]  * m[1] * m[15] + m[4]  * m[3] * m[13] + m[12] * m[1] * m[7] - m[12] * m[3] * m[5];
    inv[14] = -m[0]  * m[5] * m[14] + m[0]  * m[6] * m[13] + m[4]  * m[1] * m[14] - m[4]  * m[2] * m[13] - m[12] * m[1] * m[6] + m[12] * m[2] * m[5];
    inv[3] = -m[1] * m[6] * m[11] + m[1] * m[7] * m[10] + m[5] * m[2] * m[11] - m[5] * m[3] * m[10] - m[9] * m[2] * m[7] + m[9] * m[3] * m[6];
    inv[7] = m[0] * m[6] * m[11] - m[0] * m[7] * m[10] - m[4] * m[2] * m[11] + m[4] * m[3] * m[10] + m[8] * m[2] * m[7] - m[8] * m[3] * m[6];
    inv[11] = -m[0] * m[5] * m[11] + m[0] * m[7] * m[9] + m[4] * m[1] * m[11] - m[4] * m[3] * m[9] - m[8] * m[1] * m[7] + m[8] * m[3] * m[5];
    inv[15] = m[0] * m[5] * m[10] - m[0] * m[6] * m[9] - m[4] * m[1] * m[10] + m[4] * m[2] * m[9] + m[8] * m[1] * m[6] - m[8] * m[2] * m[5];

    double det = m[0] * inv[0] + m[1] * inv[4] + m[2] * inv[8] + m[3] * inv[12];

    if (abs(det) < 1e-9) {
        return false;
    }

    det = 1.0 / det;

    #pragma unroll
    for (int i = 0; i < 4; i++) {
        #pragma unroll
        for (int j = 0; j < 4; j++) {
            inv_out[i][j] = inv[i * 4 + j] * det;
        }
    }

    return true;
}

__global__ void best_local_affine_kernel(float *output, float *input, float *affine_model, int h, int w, float epsilon, int kernel_radius)
{
    int size = h * w;
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < size) {
        int x = id % w, y = id / w;

        double Mt_M[4][4] = {{1e-3, 0, 0, 0}, {0, 1e-3, 0, 0}, {0, 0, 1e-3, 0}, {0, 0, 0, 0}};
        double Mt_S[3][4] = {};
        double A[3][4] = {};

        for (int dy = -kernel_radius; dy <= kernel_radius; dy++) {
            for (int dx = -kernel_radius; dx <= kernel_radius; dx++) {
                int xx = x + dx, yy = y + dy;
                int id2 = yy * w + xx;

                if (0 <= xx && xx < w && 0 <= yy && yy < h) {
                    #pragma unroll
                    for (int i = 0; i < 4; i++) {
                        Mt_M[i][0] += input[id2 + 2 * size + i - 3] * input[id2 + 2 * size];
                        Mt_M[i][1] += input[id2 + size * (i - 2)] * input[id2 + 2 * size];
                        Mt_M[i][2] += input[id2 + size * (i - 1)] * input[id2 + 2 * size];
                        Mt_M[i][3] += input[id2 + 2 * size];
                    }

                    Mt_S[0][0] += input[id2 + 2 * size] * output[id2 + 2 * size];
                    Mt_S[0][1] += input[id2 + size] * output[id2 + 2 * size];
                    Mt_S[0][2] += input[id2] * output[id2 + 2 * size];
                    Mt_S[0][3] += output[id2 + 2 * size];

                    Mt_S[1][0] += input[id2 + 2 * size] * output[id2 + size];
                    Mt_S[1][1] += input[id2 + size] * output[id2 + size];
                    Mt_S[1][2] += input[id2] * output[id2 + size];
                    Mt_S[1][3] += output[id2 + size];

                    Mt_S[2][0] += input[id2 + 2 * size] * output[id2];
                    Mt_S[2][1] += input[id2 + size] * output[id2];
                    Mt_S[2][2] += input[id2] * output[id2];
                    Mt_S[2][3] += output[id2];
                }
            }
        }

        if (InverseMat4x4(Mt_M, Mt_M)) {
            for (int i = 0; i < 3; i++) {
                for (int j = 0; j < 4; j++) {
                    for (int k = 0; k < 4; k++) {
                        A[i][j] += Mt_M[j][k] * Mt_S[i][k];
                    }
                }
            }

            for (int i = 0; i < 3; i++) {
                for (int j = 0; j < 4; j++) {
                    affine_model[12 * id + i * 4 + j] = A[i][j];
                }
            }
        }
    }
}