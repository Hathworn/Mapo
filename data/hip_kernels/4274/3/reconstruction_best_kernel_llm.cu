#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 1e-4

__global__ void reconstruction_best_kernel(float *input, float *filtered_affine_model, float *filtered_best_output, int h, int w) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;
    
    if (id < size) {
        // Use shared memory for affine model
        __shared__ float shared_affine_model[12];
        
        // Load the affine model into shared memory
        if (threadIdx.x < 12) {
            shared_affine_model[threadIdx.x] = filtered_affine_model[id * 12 + threadIdx.x];
        }
        __syncthreads();
        
        // Compute outputs
        double out1 = input[id + 2*size] * shared_affine_model[0] +
                      input[id + size] * shared_affine_model[1] +
                      input[id] * shared_affine_model[2] +
                      shared_affine_model[3];
                      
        double out2 = input[id + 2*size] * shared_affine_model[4] +
                      input[id + size] * shared_affine_model[5] +
                      input[id] * shared_affine_model[6] +
                      shared_affine_model[7];
                      
        double out3 = input[id + 2*size] * shared_affine_model[8] +
                      input[id + size] * shared_affine_model[9] +
                      input[id] * shared_affine_model[10] +
                      shared_affine_model[11];
                      
        // Store the results
        filtered_best_output[id] = out1;
        filtered_best_output[id + size] = out2;
        filtered_best_output[id + 2*size] = out3;
    }
}