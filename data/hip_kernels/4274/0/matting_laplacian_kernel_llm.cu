#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 1e-4

__global__ void matting_laplacian_kernel( float *input, float *grad, int h, int w, int *CSR_rowIdx, int *CSR_colIdx, float *CSR_val, int N )
{
    int size = h * w;
    int _id = blockIdx.x * blockDim.x + threadIdx.x;

    if (_id < size) {
        int x = _id % w, y = _id / w;
        int id = x * h + y;

        // Use binary search to find the starting index
        int start = 0;
        int end = N - 1;
        int index = -1;
        while (start <= end) {
            int mid = (start + end) / 2;
            int rowIdx = (CSR_rowIdx[mid]) - 1;

            if (rowIdx == id) {
                index = mid;
                break;
            }
            if (rowIdx > id) {
                end = mid - 1;
            } else {
                start = mid + 1;
            }
        }

        if (index != -1) {
            // Determine the range of column indices for this row
            int index_s = index;
            int index_e = index;
            while (index_s >= 0 && (CSR_rowIdx[index_s] - 1) == id) index_s--;
            while (index_e < N && (CSR_rowIdx[index_e] - 1) == id) index_e++;

            // Sum up the values for this row
            for (int i = index_s + 1; i < index_e; i++) {
                int _colIdx = CSR_colIdx[i] - 1;
                float val  = CSR_val[i];

                int _x = _colIdx / h, _y = _colIdx % h;
                int colIdx = _y * w + _x;

                // Accumulate gradient using shared input values
                float inputCol = input[colIdx];
                grad[_id]         += 2 * val * inputCol;
                grad[_id + size]  += 2 * val * input[colIdx + size];
                grad[_id + 2*size] += 2 * val * input[colIdx + 2*size];
            }
        }
    }
}