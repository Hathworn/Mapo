#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate unique global position within grid
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread operates within valid boundary
    if (i < h && j < w) {
        // Compute 1D position for accessing image and normalization_factor arrays
        const int pos = i * s + j;

        // Retrieve normalization factor for current position
        float scale = normalization_factor[pos];

        // Compute and apply inverse scale factor
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

        image[pos] *= invScale;
    }
}