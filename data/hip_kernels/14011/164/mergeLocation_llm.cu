#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Compute point index
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if point index is in range
    if (ptidx < npoints)
    {
        // Use register variables for temporary storage
        short2 loc = loc_[ptidx];

        // Store scaled location
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}