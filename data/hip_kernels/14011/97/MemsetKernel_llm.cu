#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate the global thread index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if thread is out of bounds
    if (i < h && j < w) {
        // Calculate the position in the image
        const int pos = i * w + j;
        // Set the value
        image[pos] = value;
    }
}