#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    // Calculate global index for thread
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Exit if indices are out of bounds
    if (i >= h || j >= w) return;

    // Calculate row offsets
    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate target coordinates with time scaling applied
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // Calculate pixel and fractional parts using modff
    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    // Convert to integer coordinates
    int tx = (int)px;
    int ty = (int)py;

    // Precompute source pixel value
    float value = src[image_row_offset + j];
    float weight;

    // Bottom right corner
    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        weight = dx * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Bottom left corner
    tx -= 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        weight = (1.0f - dx) * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Upper left corner
    ty -= 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Upper right corner
    tx += 1;
    if (tx < w && tx >= 0 && ty < h && ty >= 0) {
        weight = dx * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}