#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeGlobalMemoryCoalesced(float *output, int size, int repeat)
{
    int gid = threadIdx.x + (blockDim.x * blockIdx.x);
    int offsets[16] = {0, 32768, 65536, 98304, 131072, 163840, 196608, 229376, 262144, 294912, 327680, 360448, 393216, 425984, 458752, 491520};
    
    for (int j = 0; j < repeat; ++j) 
    {
        #pragma unroll  // Unroll loop to improve performance
        for (int i = 0; i < 16; ++i) 
        {
            output[(gid + offsets[i]) & (size - 1)] = gid;
        }
        gid = (gid + 524288) & (size - 1); // Update gid for next iteration
    }
}