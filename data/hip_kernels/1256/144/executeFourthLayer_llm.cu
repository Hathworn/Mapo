#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void executeFourthLayer(float *Layer4_Neurons_GPU, float *Layer4_Weights_GPU, float *Layer5_Neurons_GPU)
{
    int blockID = blockIdx.x;
    int weightBegin = blockID * 101;

    // Initialize result with the bias term
    float result = Layer4_Weights_GPU[weightBegin];
    ++weightBegin;

    // Use shared memory to cache Layer4_Neurons_GPU values for improved memory access speed
    __shared__ float sharedNeurons[100];
    int neuronIdx = threadIdx.x;
    if (neuronIdx < 100) {
        sharedNeurons[neuronIdx] = Layer4_Neurons_GPU[neuronIdx + (100 * blockIdx.y)];
    }
    __syncthreads();

    // Parallel reduction over neurons and weights
    float partialSum = 0;
    for (int i = threadIdx.x; i < 100; i += blockDim.x) {
        partialSum += sharedNeurons[i] * Layer4_Weights_GPU[weightBegin + i];
    }

    // Aggregate partial sums from all threads in the block
    atomicAdd(&result, partialSum);

    // Ensure all reduction operations are completed
    __syncthreads();

    // Apply activation function
    if (neuronIdx == 0) {
        result = (1.7159f * tanhf(0.66666667f * result));
        Layer5_Neurons_GPU[blockID + (10 * blockIdx.y)] = result;
    }
}