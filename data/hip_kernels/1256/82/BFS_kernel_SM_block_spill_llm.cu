#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_SM_block_spill(volatile unsigned int *frontier, volatile unsigned int *frontier2, unsigned int frontier_len, volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, volatile unsigned int *frontier_length, const unsigned int max_mem) {
    // Calculate the global thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within the frontier length
    if (tid < frontier_len) {
        unsigned int vertex = frontier[tid];
        
        // Iterate through edges connected to the vertex
        unsigned int startEdge = edgeArray[vertex];
        unsigned int endEdge = (vertex + 1 < numVertices) ? edgeArray[vertex + 1] : numEdges;
        
        for (unsigned int edge = startEdge; edge < endEdge; ++edge) {
            unsigned int neighbor = edgeArrayAux[edge];
            
            // If the neighbor has not been visited, mark it visited
            if (atomicExch(&visited[neighbor], 1) == 0) {
                cost[neighbor] = cost[vertex] + 1; // Update cost
                unsigned int index = atomicAdd(frontier_length, 1); // Update new frontier length
                
                // Check if we have enough memory
                if (index < max_mem) {
                    frontier2[index] = neighbor; // Add neighbor to new frontier
                }
            }
        }
    }
}