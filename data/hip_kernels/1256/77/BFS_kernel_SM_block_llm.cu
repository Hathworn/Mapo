#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_SM_block(volatile unsigned int *frontier, volatile unsigned int *frontier2,
                                    unsigned int frontier_len, volatile unsigned int *cost,
                                    volatile int *visited, unsigned int *edgeArray,
                                    unsigned int *edgeArrayAux, unsigned int numVertices,
                                    unsigned int numEdges, volatile unsigned int *frontier_length,
                                    unsigned int num_p_per_mp, unsigned int w_q_size) {
    // Calculate the global thread index
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Handle out-of-bounds threads
    if (tid >= frontier_len) return;

    // Fetch the vertex from the frontier
    unsigned int vertex = frontier[tid];

    // Iterate over all outgoing edges of the vertex
    for (unsigned int edge = edgeArrayAux[vertex]; edge < edgeArrayAux[vertex + 1]; edge++) {
        unsigned int neighbor = edgeArray[edge];

        // Proceed if the neighbor hasn't been visited
        if (atomicExch(&visited[neighbor], 1) == 0) {
            // Update the cost and add the neighbor to the new frontier
            cost[neighbor] = cost[vertex] + 1;
            unsigned int idx = atomicAdd(frontier_length, 1);
            
            // Ensure the new index is within bounds
            if (idx < w_q_size) {
                frontier2[idx] = neighbor;
            }
        }
    }
}