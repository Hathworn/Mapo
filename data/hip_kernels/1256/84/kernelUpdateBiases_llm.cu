#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelUpdateBiases(float *nabla_b, float *biases, float eta, float mini_batch_size) {
    // Calculate learning rate once per block
    float rate = eta / mini_batch_size;
    
    // Use blockDim.x to support larger workloads
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if idx is within bounds
    if (idx < blockDim.x) {
        biases[idx] -= rate * nabla_b[idx];
    }
}