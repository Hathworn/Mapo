#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelFeedForward3(float *zs, float *biases, int b_off, float *activations) {
    // Use a single computation for the thread's index:
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load bias and update zs in a separate step for better readability:
    float bias = biases[b_off + threadIdx.x];
    zs[idx] += bias;

    // Optimize activation computation with single expf call:
    float z = zs[idx];
    activations[idx] = 1.0f / (1.0f + expf(-z));
}