#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeGlobalMemoryUnit(float *output, int size, int repeat)
{
    int gid = threadIdx.x + (blockDim.x * blockIdx.x);
    int s = gid * 512;

    // Unroll the loop to minimize loop overhead and achieve better performance
    for (int j = 0; j < repeat; ++j)
    {
        int baseIndex = s & (size - 1); // Calculate once to reuse
        #pragma unroll
        for (int i = 0; i < 16; ++i)
        {
            output[(baseIndex + i) & (size - 1)] = gid;
        }
        s += 16;
    }
}