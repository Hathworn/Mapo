#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_one_block(volatile unsigned int *frontier, unsigned int frontier_len, 
                                     volatile unsigned int *cost, volatile int *visited, 
                                     unsigned int *edgeArray, unsigned int *edgeArrayAux, 
                                     unsigned int numVertices, unsigned int numEdges, 
                                     volatile unsigned int *frontier_length, unsigned int num_p_per_mp, 
                                     unsigned int w_q_size) {
    // Calculate global thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit if thread ID exceeds frontier length
    if (tid >= frontier_len) {
        return;
    }

    // Load data into shared memory for faster access
    __shared__ unsigned int local_frontier[256];
    local_frontier[threadIdx.x] = frontier[tid];
    __syncthreads();  // Ensure all threads have loaded data into shared memory

    // Process edges
    for (unsigned int i = 0; i < numEdges; ++i) {
        if (local_frontier[threadIdx.x] == edgeArray[i]) {
            unsigned int neighbor = edgeArrayAux[i];
            if (!visited[neighbor]) {
                if (atomicCAS(&visited[neighbor], 0, 1) == 0) {
                    cost[neighbor] = cost[local_frontier[threadIdx.x]] + 1;
                    unsigned int pos = atomicAdd((unsigned int*)frontier_length, 1);
                    // Make sure we do not exceed the work queue size
                    if (pos < w_q_size) {
                        frontier[pos] = neighbor;
                    }
                }
            }
        }
    }
}