#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readGlobalMemoryUnit(float *data, float *output, int size, int repeat)
{
    int gid = threadIdx.x + (blockDim.x * blockIdx.x);
    float sum = 0;
    int s = gid * 512;

    // Use a loop to reduce code repetition
    for (int j = 0 ; j < repeat ; ++j)
    {
        #pragma unroll
        for (int i = 0; i < 16; ++i)
        {
            sum += data[(s + i) & (size - 1)];
        }
        s = (s + 16) & (size - 1);
    }

    output[gid] = sum;
}