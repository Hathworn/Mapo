#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_warp( unsigned int *levels, unsigned int *edgeArray, unsigned int *edgeArrayAux, int W_SZ, int CHUNK_SZ, unsigned int numVertices, int curr, int *flag)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate warp id
    int warpId = tid / W_SZ;

    // Early return if warpId is out of bounds
    if (warpId >= CHUNK_SZ) return;

    // Loop through vertices in chunks
    for (int i = warpId; i < numVertices; i += CHUNK_SZ)
    {
        // Check if the current level is the same as 'curr'
        if (levels[i] == curr)
        {
            // Loop through the edges for the current vertex
            for (unsigned int j = edgeArray[i]; j < edgeArray[i + 1]; j++)
            {
                unsigned int neighbor = edgeArrayAux[j];
                
                // Update levels with a new level and set flag
                if (levels[neighbor] == UINT_MAX)
                {
                    levels[neighbor] = curr + 1;
                    *flag = 1;
                }
            }
        }
    }
}