#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BFS_kernel_one_block_spill(volatile unsigned int *frontier, unsigned int frontier_len, volatile unsigned int *cost, volatile int *visited, unsigned int *edgeArray, unsigned int *edgeArrayAux, unsigned int numVertices, unsigned int numEdges, volatile unsigned int *frontier_length, const unsigned int max_mem) {
    // Calculate thread and block indices
    unsigned int tid = threadIdx.x;
    unsigned int index = blockIdx.x * blockDim.x + tid;

    // Ensure memory access is within bounds
    if (index < frontier_len) {
        unsigned int node = frontier[index];
        
        // Load node and its properties from global memory to registers
        unsigned int start = edgeArray[node];
        unsigned int end = edgeArray[node + 1];
        
        // Iterate over neighbors
        for (unsigned int edge = start; edge < end; ++edge) {
            unsigned int neighbor = edgeArrayAux[edge];
            
            // Check and update visited status
            if (atomicCAS((int*)&visited[neighbor], 0, 1) == 0) {
                cost[neighbor] = cost[node] + 1; // Update cost
                unsigned int pos = atomicAdd((unsigned int*)frontier_length, 1);
                
                // Ensure the new frontier fits within max memory and update
                if (pos < max_mem) {
                    frontier[pos] = neighbor;
                }
            }
        }
    }
}