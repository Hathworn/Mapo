#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hotspotOpt1(float *p, float* tIn, float *tOut, float sdc, int nx, int ny, int nz, float ce, float cw, float cn, float cs, float ct, float cb, float cc)
{
    float amb_temp = 80.0;

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i >= nx || j >= ny) return; // Early exit for out-of-bound threads

    int c = i + j * nx;
    int xy = nx * ny;

    int W = max(0, i - 1) + j * nx;         // Avoid condition check using max
    int E = min(nx - 1, i + 1) + j * nx;    // Avoid condition check using min
    int N = i + max(0, j - 1) * nx;         // Avoid condition check using max
    int S = i + min(ny - 1, j + 1) * nx;    // Avoid condition check using min

    float temp1, temp2, temp3;
    temp1 = temp2 = tIn[c];
    temp3 = tIn[c+xy];
    tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
              + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
    c += xy;
    W += xy;
    E += xy;
    N += xy;
    S += xy;

    for (int k = 1; k < nz-1; ++k) {
        temp1 = temp2;
        temp2 = temp3;
        temp3 = tIn[c+xy];
        tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
                  + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
        c += xy;
        W += xy;
        E += xy;
        N += xy;
        S += xy;
    }
    temp1 = temp2;
    temp2 = temp3;
    tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
              + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
}