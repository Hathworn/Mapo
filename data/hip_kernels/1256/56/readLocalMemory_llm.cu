#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readLocalMemory(const float *data, float *output, int size, int repeat)
{
    int tid = threadIdx.x;
    int localSize = blockDim.x;
    int grpid = blockIdx.x;
    int gid = tid + (localSize * grpid);
    int litems = 2048 / localSize;
    int goffset = localSize * grpid + tid * litems;

    __shared__ float lbuf[2048];

    // Load data only once
    for (int j = 0; j < litems && j < (size - goffset); ++j)
        lbuf[tid * litems + j] = data[goffset + j];
    
    __syncthreads();

    float sum = 0;
    int s = tid * litems;

    // Reduce redundant calculations
    for (int j = 0; j < repeat; ++j)
    {
        #pragma unroll
        for(int k = 0; k < 16; ++k)
            sum += lbuf[(s + k) & 2047];
        s = (s + 16) & 2047;
    }
    
    output[gid] = sum;
}