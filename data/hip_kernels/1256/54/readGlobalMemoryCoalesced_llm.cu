#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readGlobalMemoryCoalesced(float *data, float *output, int size, int repeat)
{
    int gid = threadIdx.x + blockDim.x * blockIdx.x;
    float sum = 0;
    int s = gid;
    #pragma unroll // Loop unrolling for better performance
    for (int j = 0; j < repeat; ++j)
    {
        #pragma unroll // Unroll inner calculations
        for (int k = 0; k < 16; ++k)
        {
            sum += data[(s + k * 32768) & (size - 1)];
        }
        s = (s + 524288) & (size - 1);
    }
    output[gid] = sum;
}