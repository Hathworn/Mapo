#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFeedForward1b(float *zs, int bound, float *weights, int w_off, float *activations) {

    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    // Initialize zs for this thread
    float sum = 0.0f;
    
    // Use shared memory for activations to improve memory access efficiency
    __shared__ float shared_activations[256]; // Adjust size according to block size
    
    // Load activations to shared memory
    if (threadIdx.x < bound) {
        shared_activations[threadIdx.x] = activations[(blockIdx.x * bound) + threadIdx.x];
    }
    __syncthreads();
    
    // Computation using shared memory
    for (int i = 0; i < bound; i++) {
        sum += weights[w_off + (threadIdx.x * bound) + i] * shared_activations[i];
    }
    
    // Store result in global memory
    zs[idx] = sum;
}