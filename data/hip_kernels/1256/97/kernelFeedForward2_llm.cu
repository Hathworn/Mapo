#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFeedForward2(float *zs, float *biases, int b_off, float *activations) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    zs[idx] += biases[b_off + idx]; // Perform bias addition
    activations[idx] = 1.0f / (1.0f + expf(-zs[idx])); // Calculate activation using sigmoid function
}