#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelBackprop3b(float *delta_nabla_b, int b_off, float *zs) {
    // Cache index value to avoid redundant calculations
    int idx = b_off + threadIdx.x;
    float sigmoid = 1.0 / (1.0 + expf(-zs[threadIdx.x])); // Calculate sigmoid once
    delta_nabla_b[idx] *= sigmoid * (1.0 - sigmoid);
}