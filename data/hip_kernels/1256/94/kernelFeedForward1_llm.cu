#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelFeedForward1(float *zs, int bound2, float *weights, int w_off, float *activations1) {
    int tx = threadIdx.x;

    // Initialize zs to 0 for the current thread
    float z_value = 0.0f;

    // Use a pointer for weights to avoid calculating the offset multiple times
    float* weight_ptr = weights + w_off + (tx * bound2);

    // Accumulate the weighted sum
    for (int i = 0; i < bound2; i++) {
        z_value += weight_ptr[i] * activations1[i];
    }

    // Store the result in zs
    zs[tx] = z_value;
}