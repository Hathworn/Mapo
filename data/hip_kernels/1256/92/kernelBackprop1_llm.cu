#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelBackprop1(float *delta_nabla_w, int w_off, float *activations, float *delta_nabla_b, int b_off) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Compute global index

    // Efficient memory access pattern
    delta_nabla_w[w_off + idx] = activations[threadIdx.x] * delta_nabla_b[b_off + blockIdx.x];
}