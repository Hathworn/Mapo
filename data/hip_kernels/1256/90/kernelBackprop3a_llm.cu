#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelBackprop3a(float *delta_nabla_b, int b_off, int bound, int b_off_old, float *weights, int w_off_old) {
    // Calculate the index for the current thread
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Initialize shared memory for the reduction
    extern __shared__ float shared_mem[];

    // Initialize delta_nabla_b for the current thread
    float sum = 0.0f;

    // Use a loop unrolling technique for performance improvement
    for (int j = 0; j < bound; j += 4) {
        if (j < bound) sum += delta_nabla_b[b_off_old + j] * weights[w_off_old + (j * blockDim.x) + threadIdx.x];
        if (j + 1 < bound) sum += delta_nabla_b[b_off_old + j + 1] * weights[w_off_old + ((j + 1) * blockDim.x) + threadIdx.x];
        if (j + 2 < bound) sum += delta_nabla_b[b_off_old + j + 2] * weights[w_off_old + ((j + 2) * blockDim.x) + threadIdx.x];
        if (j + 3 < bound) sum += delta_nabla_b[b_off_old + j + 3] * weights[w_off_old + ((j + 3) * blockDim.x) + threadIdx.x];
    }

    // Write result back to global memory
    delta_nabla_b[b_off + idx] = sum;
}