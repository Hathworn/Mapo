#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void timeTest1(int *a) {
    *a += 5; // Simplified computation
}

__global__ void timeTest() {
    int t_index = threadIdx.x + (blockIdx.x * blockDim.x);

    if (t_index < SIZE) {
        int a = 0;
        #pragma unroll  // Unroll loop for performance
        for (int i = 0; i < 10000000; i++) {
            timeTest1(&a);
        }
    }
}