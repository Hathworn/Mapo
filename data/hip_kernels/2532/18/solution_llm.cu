#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void solution(float *f, float lambda, int nx, int ny)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds before computation to avoid unnecessary index calculation
    if (x < nx && y < ny) {
        int idx = x + y * nx;
        f[idx] = -f[idx] * lambda;
    }
}