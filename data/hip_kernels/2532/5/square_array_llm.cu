#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride-based loop for better occupancy, handling more data in case of large arrays
    int stride = gridDim.x * blockDim.x;
    for (int i = idx; i < N; i += stride) {
        a[i] = a[i] * a[i];
    }
}