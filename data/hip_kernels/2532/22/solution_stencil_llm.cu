#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void solution_stencil(float *zx, float *zy, float *g, float lambda, int nx, int ny)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;

    if (px < nx && py < ny)
    {
        int idx = px + py * nx;
        float DIVZ = 0.0f;

        // Optimize divergence computation with fewer condition checks
        if (px < (nx - 1)) DIVZ += zx[idx];
        else if (px > 0) DIVZ -= zx[idx - 1];
        
        if (py < (ny - 1)) DIVZ += zy[idx];
        else if (py > 0) DIVZ -= zy[idx - nx];

        // Update g with computed divergence
        g[idx] = -DIVZ * lambda + g[idx];
    }
}