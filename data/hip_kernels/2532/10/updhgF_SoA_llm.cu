#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updhgF_SoA(float *f, float *z1, float *z2, float *g, float tf, float invlambda, int nx, int ny)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate global thread index
    int idx = px + py * nx;

    if (px < nx && py < ny)  // Ensure within bounds
    {
        float DIVZ = 0.0f;

        // Use shared memory to enhance data locality and reduce global memory access.
        // Assuming blockDim.x * blockDim.y <= shared memory limit.
        __shared__ float shared_z1[1024], shared_z2[1024];

        int localIdx = threadIdx.y * blockDim.x + threadIdx.x;
        shared_z1[localIdx] = z1[idx];
        shared_z2[localIdx] = z2[idx];

        __syncthreads();

        // Compute the divergence using shared data
        if (px < (nx - 1)) DIVZ += shared_z1[localIdx];
        if (px > 0)        DIVZ -= shared_z1[localIdx - 1];

        if (py < (ny - 1)) DIVZ += shared_z2[localIdx];
        if (py > 0)        DIVZ -= shared_z2[localIdx - blockDim.x];

        __syncthreads();

        // Update f
        f[idx] = (1 - tf) * f[idx] + tf * (g[idx] + invlambda * DIVZ);
    }
}
```
