#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void solution2(float *f, float lambda, int nx, int ny)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensuring index is valid before calculating idx
    if (x < nx && y < ny) {
        int idx = x + y * nx;
        f[idx] = -f[idx] * lambda;
    }
}