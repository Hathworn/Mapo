#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fupdate_SoA(float *f, float *z1, float *z2, float *g, float invlambda, int nx, int ny)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = px + py * nx;
    
    if (px < nx && py < ny)
    {
        // Load Z1 and Z2 values once to reduce global memory access
        float Z1c = z1[idx];
        float Z2c = z2[idx];
        float Z1l = (px > 0) ? z1[idx - 1] : 0.0f;
        float Z2d = (py > 0) ? z2[idx - nx] : 0.0f;

        // compute the divergence using conditional reduction
        float DIVZ = Z1c - Z1l + Z2c - Z2d;

        // update f
        f[idx] = DIVZ - g[idx] * invlambda;
    }
}