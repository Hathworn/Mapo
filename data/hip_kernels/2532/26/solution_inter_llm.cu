#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void solution_inter(float *z, float *g, float lambda, int nx, int ny)
{
    // Calculate global thread ID
    int idx = threadIdx.x + blockIdx.x * blockDim.x + 
              (threadIdx.y + blockIdx.y * blockDim.y) * nx;

    // Ensure bounds check within the grid limits
    if (idx < nx * ny) {
        g[idx] = -z[3 * idx + 2] * lambda;
    }
}