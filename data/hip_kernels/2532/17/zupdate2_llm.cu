#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void zupdate2(float *z, float *f, float tau, int nx, int ny)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = px + py * nx;

    if (px < nx && py < ny)
    {
        float a = 0.0f, b = 0.0f, t, fc = f[idx];

        // Use conditional operator for cleaner gradient computation
        a = (px < nx - 1) ? f[idx + 1] - fc : 0.0f;
        b = (py < ny - 1) ? f[idx + nx] - fc : 0.0f;

        // Optimize division by pre-computing reciprocal
        float norm = sqrtf(a * a + b * b);
        float tau_norm_recip = 1.0f / (1.0f + tau * norm);

        // Update z
        z[2 * idx] = (z[2 * idx] + tau * a) * tau_norm_recip;
        z[2 * idx + 1] = (z[2 * idx + 1] + tau * b) * tau_norm_recip;
    }
}