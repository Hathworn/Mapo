#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void divergenceL(float *v, float *d, int nx, int ny)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    // Calculate index only once
    int idx = px + py * nx;

    // Optimize by adding boundary checks before performing operations
    if (px < nx && py < ny)
    {
        float AX = 0;

        // Utilize conditional operator to reduce branching
        AX += (px < (nx - 1)) ? v[2 * idx + 0] : 0;
        AX -= (px > 0) ? v[2 * (idx - 1) + 0] : 0;

        AX += (py < (ny - 1)) ? v[2 * idx + 1] : 0;
        AX -= (py > 0) ? v[2 * (idx - nx) + 1] : 0;

        d[idx] = AX; // Direct assignment without unnecessary check
    }
}