#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void zupdate_inter(float *z, float tau, int nx, int ny)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure thread is within bounds before any calculations
    if (px >= nx || py >= ny) return;

    int idx = px + py * nx;
    float a = 0.0f, b = 0.0f, t;

    // Compute gradients
    if (px < nx - 1) a = z[3 * (idx + 1) + 2] - z[3 * idx + 2];
    if (py < ny - 1) b = z[3 * (idx + nx) + 2] - z[3 * idx + 2];

    // Pre-compute common expressions
    float magnitude = sqrtf(a * a + b * b);
    t = 1.0f / (1.0f + tau * magnitude);

    // Update z with computed values
    z[3 * idx + 0] = (z[3 * idx + 0] + tau * a) * t;
    z[3 * idx + 1] = (z[3 * idx + 1] + tau * b) * t;
}