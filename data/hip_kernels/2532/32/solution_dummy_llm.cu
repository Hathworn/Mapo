#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void solution_dummy(float *f, float lambda, int nx, int ny)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = x + y * nx;

    // Ensure thread operates within bounds first
    if (x < nx && y < ny) {
        // Load from global memory to register for faster access
        float temp = f[idx];
        f[idx] = -temp * lambda;
    }
}