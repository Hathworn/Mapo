#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updF_SoA(float *f, float *z1, float *z2, float *g, float tf, float lambda, int nx, int ny)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Pre-calculate indices
    int idx = px + py * nx;
    if (px < nx && py < ny)
    {
        float DIVZ = 0.0f;
        if (px < nx - 1) DIVZ += z1[idx];
        if (px > 0)      DIVZ -= z1[idx - 1];
        
        if (py < ny - 1) DIVZ += z2[idx];
        if (py > 0)      DIVZ -= z2[idx - nx];
        
        // Optimize division by precomputing reciprocal
        float invDenom = 1.0f / (1.0f + tf * lambda);
        f[idx] = (f[idx] + tf * DIVZ + tf * lambda * g[idx]) * invDenom; // Simplified update
    }
}