#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void zupdate(float *z, float *z0, float tau, int nx, int ny)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = x + y * nx;

    if (x < nx && y < ny)
    {
        // Unroll loop for better performance
        float a = z[2 * idx];
        float b = z[2 * idx + 1];
        float scale = 1.0f / (1.0f + tau * sqrtf(a * a + b * b));

        z[2 * idx] = (z0[2 * idx] + tau * a) * scale;
        z[2 * idx + 1] = (z0[2 * idx + 1] + tau * b) * scale;
    }
}