#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sub0(float *div0, float *div, float *g, float lambda, int nx, int ny)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < nx && y < ny)
    {
        int idx = x + y * nx; // Calculate index only for valid threads
        div[idx] = div0[idx] - g[idx] / lambda; // Perform the calculation
    }
}