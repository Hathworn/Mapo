#include "hip/hip_runtime.h"
```cuda
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divergence(float *v, float *d, int nx, int ny)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread only processes valid indices
    if (px < nx && py < ny)
    {
        int idx = px + py * nx;
        float AX = 0;
        
        // Unroll operations to reduce calculations
        if (px < nx - 1) AX += v[2 * idx];
        if (px > 0)      AX -= v[2 * (idx - 1)];

        if (py < ny - 1) AX += v[2 * idx + 1];
        if (py > 0)      AX -= v[2 * (idx - nx) + 1];

        // Store result
        d[idx] = AX;
    }
}