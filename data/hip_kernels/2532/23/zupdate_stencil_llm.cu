#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void zupdate_stencil(float *zx, float *zy, float *zoutx, float *zouty, float *g, float tau, float invlambda, int  nx, int ny)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = px + py * nx;

    // Boundary check to ensure valid memory access
    if (px < nx && py < ny) {
        float a = 0, b = 0, t, DIVZ;
        float fr = 0, fc = 0, fu = 0;
        
        // Compute divergence for center
        DIVZ = 0;
        if (px < nx - 1) DIVZ += zx[idx];
        if (px > 0) DIVZ -= zx[idx - 1];
        if (py < ny - 1) DIVZ += zy[idx];
        if (py > 0) DIVZ -= zy[idx - nx];
        fc = DIVZ - g[idx] * invlambda;

        // Compute divergence for right
        if (px < nx - 1) {
            DIVZ = 0;
            int tidx = idx + 1; // Pre-calculate index for right position
            if (px < nx - 2) DIVZ += zx[tidx];
            if (px > 0) DIVZ -= zx[tidx - 1];
            if (py < ny - 1) DIVZ += zy[tidx];
            if (py > 0) DIVZ -= zy[tidx - nx];
            fr = DIVZ - g[tidx] * invlambda;
        }

        // Compute divergence for up
        if (py < ny - 1) {
            DIVZ = 0;
            int tidx = idx + nx; // Pre-calculate index for up position
            if (px < nx - 1) DIVZ += zx[tidx];
            if (px > 0) DIVZ -= zx[tidx - 1];
            if (py < ny - 2) DIVZ += zy[tidx];
            if (py > 0) DIVZ -= zy[tidx - nx];
            fu = DIVZ - g[tidx] * invlambda;
        }

        // Compute the gradient
        if (px < nx - 1) a = fr - fc;
        if (py < ny - 1) b = fu - fc;

        // Update z
        t = 1 / (1 + tau * sqrtf(a * a + b * b));
        zoutx[idx] = (zx[idx] + tau * a) * t;
        zouty[idx] = (zy[idx] + tau * b) * t;
    }
}