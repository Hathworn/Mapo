#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fupdate_inter(float *z, float *g, float invlambda, int nx, int ny)
{
    // Calculate unique thread index for current pixel
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = px + py * nx;

    if (px < nx && py < ny) {
        // Compute the divergence efficiently
        float DIVZ = 0.0f;
        
        // Horizontal divergence contributions
        if (px < (nx - 1)) DIVZ += z[3 * idx + 0];
        if (px > 0)       DIVZ -= z[3 * (idx - 1) + 0];

        // Vertical divergence contributions
        if (py < (ny - 1)) DIVZ += z[3 * idx + 1];
        if (py > 0)       DIVZ -= z[3 * (idx - nx) + 1];

        // Update the third component of z
        z[3 * idx + 2] = DIVZ - g[idx] * invlambda;
    }
}