#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gradientL(float *u, float *g, int nx, int ny)
{
    // Calculate pixel indices
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;

    // Flatten 2D index to 1D
    int idx = px + py * nx;

    // Ensure computation only within valid range
    if (px < nx && py < ny)
    {
        // Initialize gradient components to zero
        g[2 * idx] = 0;
        g[2 * idx + 1] = 0;

        // Compute x-direction gradient if not at boundary
        if (px < (nx - 1)) 
            g[2 * idx] = u[idx + 1] - u[idx];
        
        // Compute y-direction gradient if not at boundary
        if (py < (ny - 1)) 
            g[2 * idx + 1] = u[idx + nx] - u[idx];
    }
}