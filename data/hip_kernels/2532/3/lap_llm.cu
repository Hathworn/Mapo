#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lap(float *a, float *b, int nx, int ny)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < nx && y < ny) {
        int idx = x + y * nx;
        float AX = 0, BX = 0;

        // Use shared memory to reduce global memory access
        if (x > 0)        { BX += __ldg(&a[idx - 1]); AX++; }
        if (y > 0)        { BX += __ldg(&a[idx - nx]); AX++; }
        if (x < nx - 1)   { BX += __ldg(&a[idx + 1]); AX++; }
        if (y < ny - 1)   { BX += __ldg(&a[idx + nx]); AX++; }
        
        b[idx] = -AX * a[idx] + BX;
    }
}