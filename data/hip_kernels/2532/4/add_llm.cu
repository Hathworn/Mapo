#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(float *a, float *b, int nx, int ny)
{
    // Calculate the linear index directly
    int idx = blockIdx.x * blockDim.x + threadIdx.x 
            + (blockIdx.y * blockDim.y + threadIdx.y) * nx;
            
    // Check within bounds and perform computation
    if (blockIdx.x * blockDim.x + threadIdx.x < nx && blockIdx.y * blockDim.y + threadIdx.y < ny) 
    {
        b[idx] += a[idx] * 0.125f;  // Use float literal for precise computation
    }
}