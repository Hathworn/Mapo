#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updZ(float *z, float *f, float tz, float beta, int nx, int ny)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = px + py * nx;

    if (px < nx && py < ny)
    {
        float a = 0.0f, b = 0.0f;

        // Compute the gradient only if within bounds
        if (px < (nx - 1)) a = f[idx + 1] - f[idx];
        if (py < (ny - 1)) b = f[idx + nx] - f[idx];

        // Update z using the gradient and normalize
        a = z[2 * idx] + tz * a;
        b = z[2 * idx + 1] + tz * b;

        float t = sqrtf(beta + a * a + b * b);
        t = (t < 1.0f) ? 1.0f : 1.0f / t;

        z[2 * idx] = a * t;
        z[2 * idx + 1] = b * t;
    }
}