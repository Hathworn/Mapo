#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updhgZ_SoA(float *z1, float *z2, float *f, float tz, float lambda, int nx, int ny)
{
    // Cache block and thread indices
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = px + py * nx;

    if (px < nx && py < ny)
    {
        // Use register variables for computations
        float a = 0.0f;
        float b = 0.0f;
        float fc = f[idx];

        // Avoid index recalculations
        int idx_px1 = idx + 1;
        int idx_py1 = idx + nx;

        if (px < (nx - 1)) a = f[idx_px1] - fc;
        if (py < (ny - 1)) b = f[idx_py1] - fc;

        // Update z
        a = z1[idx] + tz * lambda * a;
        b = z2[idx] + tz * lambda * b;

        // Compute projection without conditional branching
        float t = rsqrtf(a * a + b * b);
        t = fminf(t, 1.0f);

        z1[idx] = a * t;
        z2[idx] = b * t;
    }
}