#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gradient(float *u, float *g, int nx, int ny)
{
    // Calculate global index
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate linear index
    int idx = px + py * nx;

    // Check boundaries
    if (px < nx && py < ny)
    {
        // Initialize gradient components to zero
        g[2 * idx] = 0;
        g[2 * idx + 1] = 0;

        // Compute gradient along the x-direction if within bounds
        if (px < nx - 1) g[2 * idx] = u[idx + 1] - u[idx];

        // Compute gradient along the y-direction if within bounds
        if (py < ny - 1) g[2 * idx + 1] = u[idx + nx] - u[idx];
    }
}