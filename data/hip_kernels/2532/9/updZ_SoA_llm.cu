#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updZ_SoA(float *z1, float *z2, float *f, float tz, float beta, int nx, int ny)
{
    // Calculate pixel index
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = px + py * nx;

    // Ensure within valid range
    if (px < nx && py < ny)
    {
        // Pre-load f[idx] to reduce data access
        float fc = f[idx];

        // Compute the gradient conditionally
        float a = (px < nx - 1) ? f[idx + 1] - fc : 0;
        float b = (py < ny - 1) ? f[idx + nx] - fc : 0;

        // Efficiently update z
        a = z1[idx] + tz * a;
        b = z2[idx] + tz * b;

        float t = sqrtf(beta + a * a + b * b);
        t = fminf(1.0f, 1.0f / t);

        // Update z1 and z2
        z1[idx] = a * t;
        z2[idx] = b * t;
    }
}