#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void solution_SoA(float *f, float lambda, int nx, int ny)
{
    // Cache calculations in registers
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < nx && y < ny) {
        int idx = x + y * nx; // Move index calculation inside the condition
        f[idx] = -f[idx] * lambda;
    }
}