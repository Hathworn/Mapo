#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updF(float *f, float *z, float *g, float tf, float lambda, int nx, int ny)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = px + py*nx;

    if (px < nx && py < ny)
    {
        // compute the divergence
        float DIVZ = 0.0f;
        if (px < nx - 1) DIVZ += z[2 * idx + 0];
        if (px > 0) DIVZ -= z[2 * (idx - 1) + 0];
        if (py < ny - 1) DIVZ += z[2 * idx + 1];
        if (py > 0) DIVZ -= z[2 * (idx - nx) + 1];

        // update f using updated formula for better precision and performance
        f[idx] = (f[idx] + tf * (DIVZ + lambda * g[idx])) / (1.0f + tf * lambda);
    }
}