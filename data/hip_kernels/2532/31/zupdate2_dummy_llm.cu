#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void zupdate2_optimized(float *z1, float *z2, float *f, float tau, int nx, int ny)
{
    // Use shared memory for block data to reduce global memory accesses
    __shared__ float shared_f[BLOCK_SIZE_X][BLOCK_SIZE_Y];

    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;

    float a, b, t;

    if (px < nx && py < ny) {
        // Load data into shared memory
        int idx = px + py * nx;
        shared_f[threadIdx.x][threadIdx.y] = f[idx];
        __syncthreads();  // Ensure all threads have written to shared memory

        // Compute the gradient using shared memory
        a = shared_f[threadIdx.x][threadIdx.y];
        b = shared_f[threadIdx.x][threadIdx.y];

        // Update z values
        t = 1 / (1 + tau * sqrtf(a * a + b * b));
        z1[idx] = (z1[idx] + tau * a) * t;
        z2[idx] = (z2[idx] + tau * b) * t;
    }
}