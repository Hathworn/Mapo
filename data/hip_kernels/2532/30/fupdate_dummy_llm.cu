#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void fupdate_dummy(float *f, float *z1, float *z2, float *g, float invlambda, int nx, int ny)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = px + py * nx;

    if (px < nx && py < ny)
    {
        // Use local variables to store reusable values
        float Z1c = z1[idx];
        float Z2c = z2[idx];
        float gVal = g[idx];

        // Compute divergence using conditional operators for cleaner code
        float DIVZ = ((px < nx - 1) ? Z1c : 0) + ((py < ny - 1) ? Z2c : 0);

        // Update f
        f[idx] = DIVZ - gVal * invlambda;
    }
}