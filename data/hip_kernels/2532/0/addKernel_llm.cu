#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate the global index for each thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we don't access out-of-bounds memory
    if (i < blockDim.x * gridDim.x)
    {
        c[i] = a[i] + b[i];
    }
}