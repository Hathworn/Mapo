#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void zupdate2_SoA(float *z1, float *z2, float *f, float tau, int nx, int ny)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;

    // Use shared memory for improved access speed
    __shared__ float s_f[32][32];  

    // Check for boundary conditions
    if (px < nx && py < ny)
    {
        int idx = px + py * nx;

        // Load the shared memory
        s_f[threadIdx.y][threadIdx.x] = f[idx];
        __syncthreads();

        float a = 0.0f, b = 0.0f;

        int rightIdx = (px < nx - 1) ? threadIdx.x + 1 : threadIdx.x;
        int upIdx = (py < ny - 1) ? threadIdx.y + 1 : threadIdx.y;

        // Use shared memory values for computation
        if (px < (nx - 1)) a = s_f[threadIdx.y][rightIdx] - s_f[threadIdx.y][threadIdx.x];
        if (py < (ny - 1)) b = s_f[upIdx][threadIdx.x] - s_f[threadIdx.y][threadIdx.x];

        // Efficient computation of sqrt and update z values
        float norm = rsqrtf(1.0f + tau * sqrtf(a * a + b * b));
        z1[idx] = (z1[idx] + tau * a) * norm;
        z2[idx] = (z2[idx] + tau * b) * norm;
    }
}