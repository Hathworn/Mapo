#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAdd(int n, const float *A, const float *B, float* C) {
    // Calculate global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if thread index is within bounds
    if (i < n) {
        // Load A and B and compute C
        C[i] = A[i] + B[i];
    }
    
    // Introduce a delay to simulate compute time
    long long start = clock64();
    long long cycles_elapsed;
    do { cycles_elapsed = clock64() - start; } while (cycles_elapsed < 20000);
}