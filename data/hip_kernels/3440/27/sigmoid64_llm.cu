#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" 
#include "includes.h"

__global__ void sigmoid64(double* A, int size) 
{
    // Optimize index calculation and simplify block id computation
    int idx = blockIdx.x * blockDim.x * blockDim.y * blockDim.z + threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    if (idx < size) {
        // Use intrinsics for better performance
        A[idx] = 1.0 / (1.0 + exp(-A[idx]));
    }
}