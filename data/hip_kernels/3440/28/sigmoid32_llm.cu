#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sigmoid32(float* A, int size)
{
    // Optimize grid stride loop for better occupancy and scalability
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (; idx < size; idx += stride) {
        A[idx] = 1.0f / (1.0f + expf(-A[idx])); // Use expf for improved performance
    }
}