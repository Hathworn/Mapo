#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setupRandomStatesKernel(hiprandState* __restrict__ states, unsigned long long seed, int count)
{
for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < count; i += blockDim.x * gridDim.x)
{
hiprand_init(seed, i, 0, &states[i]);
}
}