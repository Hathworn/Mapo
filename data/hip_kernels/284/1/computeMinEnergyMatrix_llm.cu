#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeMinEnergyMatrix(float *energy, float *min_energy, int height, int width) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int pos = bid * blockDim.x + tid;

    extern __shared__ float shared_row_energy[];

    // Load initial row into shared memory
    if(pos < width) {
        shared_row_energy[tid] = energy[tid];
        min_energy[tid] = energy[tid];
    } else {
        return;
    }
    __syncthreads();
    
    for(int i = 1; i < height; i++) {
        float l, m, r;
        float current = energy[i * width + tid];

        // Check bounds for left, middle, right
        l = (tid > 0) ? shared_row_energy[tid - 1] : 999999999;
        m = shared_row_energy[tid];
        r = (tid < width - 1) ? shared_row_energy[tid + 1] : 999999999;

        // Calculate minimum energy
        float minimum = current + min(l, min(m, r));

        // Update shared memory and output
        shared_row_energy[tid] = minimum;
        min_energy[i * width + tid] = minimum;

        __syncthreads();
    }
}