#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void phshmem_reduce_kernel(float* d_out, const float* d_in) {
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // Load shared mem from global mem and handle boundary condition
    if (myId < gridDim.x * blockDim.x) {
        sdata[tid] = d_in[myId];
    } else {
        sdata[tid] = 0.0f;
    }
    __syncthreads();

    // Unroll the last warp for efficiency
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Perform work for the last warp without synchronization
    if (tid < 32) {
        volatile float* vsdata = sdata;
        vsdata[tid] += vsdata[tid + 32];
        vsdata[tid] += vsdata[tid + 16];
        vsdata[tid] += vsdata[tid + 8];
        vsdata[tid] += vsdata[tid + 4];
        vsdata[tid] += vsdata[tid + 2];
        vsdata[tid] += vsdata[tid + 1];
    }

    // Only thread 0 writes result for this block back to global memory
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}