#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < numElements) 
    {
        // Optimize by using per-thread registers
        float a = A[i];
        float b = B[i];
        C[i] = a + b;
    }
}