#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel with better usage of shared memory
__global__ void make_bins(float *vec, int *bin, const int num_bins, const int n, const float slope, const float intercept)
{
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

    if (xIndex < n) {
        // Load vec value to shared memory (reduce global memory access)
        __shared__ float shared_vec[BLOCK_SIZE];
        shared_vec[threadIdx.x] = vec[xIndex];
        __syncthreads();

        float temp = abs(shared_vec[threadIdx.x]);
        
        if (temp > (intercept * 0.000001)) {
            bin[xIndex] = slope * (intercept - temp);
        } else {
            bin[xIndex] = num_bins;
        }
    }
}