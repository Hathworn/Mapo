#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void threshold(float *vec, int *bin, const int k_bin, const int n)
{
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Check xIndex within bounds and that bin value exceeds k_bin
    if (xIndex < n && bin[xIndex] > k_bin)
    {
        // Set vector value to zero
        vec[xIndex] = 0.0f;
    }
}