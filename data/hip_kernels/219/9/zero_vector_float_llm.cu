#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void zero_vector_float(float *vec, const int n)
{
    // Calculate global thread index
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within bounds
    if (xIndex < n)
    {
        vec[xIndex] = 0.0f; // Set the vector element to zero
    }
}