#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gen_matvecT(float *A, float *x, float *y, const int m, const int n)
{
    // Use shared memory for y to reduce global memory accesses
    __shared__ float shared_y[1024];  // Assume 1024 threads per block (adjust size as needed)
    
    // Load y elements into shared memory
    if (threadIdx.x < m) {
        shared_y[threadIdx.x] = y[threadIdx.x];
    }
    __syncthreads();  // Ensure all y elements are loaded before proceeding

    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    if (xIndex < n) {
        float c = 0.0f;
        for (int i = 0; i < m; i++) {
            c += shared_y[i] * A[xIndex * m + i];  // Use shared memory for y[i]
        }
        x[xIndex] = c;
    }
}