#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_adjacent(int *vec, int *vec_shorter, const int n)
{
    // Calculate global index
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory for faster access within a block
    extern __shared__ int shared_vec[];

    // Load data into shared memory
    if (xIndex < 2 * n) {
        shared_vec[threadIdx.x] = vec[xIndex];
    }
    __syncthreads();

    // Perform addition if within bounds
    if (xIndex < n) {
        vec_shorter[xIndex] = shared_vec[2 * threadIdx.x] + shared_vec[2 * threadIdx.x + 1];
    }
}