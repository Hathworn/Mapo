#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void count_zero_one(float *vec, float *data, const int n)
{
    // Use shared memory to reduce global memory access
    __shared__ int shared_data[2];
    if (threadIdx.x == 0) {
        shared_data[0] = 0;
        shared_data[1] = 0;
    }
    __syncthreads();

    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Use a single atomic operation to increase parallelism
    if (xIndex < n) {
        if (vec[xIndex] == 0) {
            atomicAdd(&shared_data[0], 1);
        } else if (vec[xIndex] == 1) {
            atomicAdd(&shared_data[1], 1);
        }
    }

    __syncthreads();

    // Only one thread updates global memory
    if (threadIdx.x == 0) {
        atomicAdd(data, shared_data[0]);
        atomicAdd(data + 1, shared_data[1]);
    }
}