#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void count_bins(int *bin, int *bin_counters, const int num_bins, const int n)
{
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure valid index and bin value within bounds
    if (xIndex < n && bin[xIndex] < num_bins)
    {
        // Use atomicAdd safely
        atomicAdd(bin_counters + bin[xIndex], 1);
    }
}