#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void countRest(int *bin, int *bin_counters, const int num_bins, const int maxBin, const int n)
{
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    if (xIndex < n) {
        int binValue = bin[xIndex];
        // Combine both conditions to minimize divergence
        if (binValue >= maxBin && binValue < num_bins) {
            atomicAdd(&bin_counters[binValue], 1); // Directly use binValue instead of recalculating
        }
    }
}