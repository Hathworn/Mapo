#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void halve_bins(int *bin, const int n)
{
    // Use a stride loop to handle more elements per thread efficiently
    for (unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x; xIndex < n; xIndex += blockDim.x * gridDim.x)
    {
        bin[xIndex] /= 2;
    }
}