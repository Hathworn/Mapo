#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void threshold_and_support(float *vec, int *support, const int n, const float T)
{
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if xIndex is within bounds
    if (xIndex >= n) return;

    // Use fabs to optimize the absolute value computation
    if (fabsf(vec[xIndex]) < T) {
        vec[xIndex] = 0.0f;
        support[xIndex] = 2;
    }
}