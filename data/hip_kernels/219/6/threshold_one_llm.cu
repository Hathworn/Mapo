#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void threshold_one(float *vec, float *vec_thres, int *bin, const int k_bin, const int n)
{
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use unsigned integer comparison instead of bitwise AND for bounds check
    if (xIndex < n && bin[xIndex] <= k_bin) 
    {
        vec_thres[xIndex] = vec[xIndex];
    }
}