#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void magnitude(float *vec, const int n)
{
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use intrinsics for faster absolute value calculation
    if (xIndex < n) {
        vec[xIndex] = fabsf(vec[xIndex]);
    }
}