#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dyadicAdd(int * counter, const int length, const int shift)
{
    if (shift > 0) {
        unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
        
        // Pre-calculate adds, simplifies calculation below
        int adds = 2 * shift;
        
        // Calculate the targeted index directly
        int Index = adds * (xIndex + 1) - 1;

        // Check boundary condition to prevent out-of-bounds access
        if (Index < length) {
            // Perform addition with a direct memory reference
            counter[Index] += counter[Index - shift];
        }
    }
}