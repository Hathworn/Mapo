#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void make_and_count_bins(float *vec, int *bin, int *bin_counters, const int num_bins, const int n, const float slope, const float intercept)
{
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure the index is within bounds before proceeding
    if (xIndex < n) {
        float temp = abs(vec[xIndex]);
        int binIndex;

        // Use conditional operator for concise assignment
        if (temp > (intercept * 0.01)) {
            binIndex = max(0, __float2int_rd(slope * (intercept - temp))); // Use faster intrinsic rounding
            atomicAdd(bin_counters + binIndex, 1);
        } else {
            binIndex = __float2int_rd(slope * intercept) + 1; // Use faster intrinsic rounding
        }

        bin[xIndex] = binIndex;  // Assign binIndex to the bin array
    }
}
```
