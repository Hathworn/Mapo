#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float getAbsMax(float* d_vec, const int length)
{
    float segmentMax = 0;
    for (int jj = 0; jj < length; jj++) {
        float val = abs(d_vec[jj]);
        if (segmentMax < val) segmentMax = val;  // Simplified comparison
    }
    return segmentMax;
}

__global__ void segmentMax(float* d_vec, float* segmentMaxes, const int length, const int HighLength, const int HighSegmentLength, const int threadsHigh, const int LowSegmentLength)
{
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int startIndex, SegmentLength;

    // Simplified conditional logic
    if (xIndex >= threadsHigh || (xIndex * HighSegmentLength > HighLength && (HighLength + (xIndex - threadsHigh + 1) * LowSegmentLength) < length)) {
        startIndex = HighLength + (xIndex - threadsHigh) * LowSegmentLength;
        SegmentLength = LowSegmentLength;
    } else {
        startIndex = xIndex * HighSegmentLength;
        SegmentLength = HighSegmentLength;
    }

    // Each thread calculates its segment max
    segmentMaxes[xIndex] = getAbsMax(d_vec + startIndex, SegmentLength);
}