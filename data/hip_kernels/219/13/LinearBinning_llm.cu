#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LinearBinning(float *vec, int *bin, int *bin_counters, const int num_bins, const int MaxBin, const int n, const float slope, const float intercept)
{
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

    if (xIndex < n) {
        float temp = abs(vec[xIndex]);
        // Avoid redundant computation by computing this once
        float intercept_thresh = intercept * 0.000001f;

        if (temp > intercept_thresh) {
            // Ensure the result of max() is correctly assigned as an int
            int binValue = max(0, static_cast<int>(slope * (intercept - temp)));
            bin[xIndex] = binValue;

            // Check if binValue is within range before atomicAdd
            if (binValue < MaxBin) {
                atomicAdd(bin_counters + binValue, 1);
            }
        } else {
            bin[xIndex] = static_cast<int>(slope * intercept + 1.0f);
        }
    }
}