#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void one_vector_int(int *vec, const int n)
{
    // Calculate thread index
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Efficient boundary check and assignment
    if (xIndex < n)
    {
        vec[xIndex] = 1;
    }
}