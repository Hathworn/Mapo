#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void segCountSum_shared(int *counter, int *segcounter, const int countlength)
{
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    extern __shared__ int s_counter[];

    // Initialize shared memory only for valid indices
    if (xIndex < countlength) {
        s_counter[xIndex] = 0; // Initialize shared memory to 0
        
        // Use a single loop with stride to improve memory access pattern
        for (int jj = xIndex; jj < countlength * countlength; jj += countlength) {
            s_counter[xIndex] += segcounter[jj];
        }
        
        // Write the result back to global memory
        counter[xIndex] = s_counter[xIndex];
    }
}