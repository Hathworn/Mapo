#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void check_support(float * vec_input, float * vec, const int n, int * support_counter)
{
    // Calculate unique thread index for flattened grid
    int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Check within array bounds
    if (xIndex < n) {
        // Combine conditions to reduce branching
        bool input_non_zero = (vec_input[xIndex] != 0);
        bool vec_non_zero = (vec[xIndex] != 0);

        // Use unified branching for atomic add operations
        if (input_non_zero && vec_non_zero) {
            atomicAdd(support_counter, 1);
        } else if (!input_non_zero && !vec_non_zero) {
            atomicAdd(support_counter + 1, 1);
        }
    }
}