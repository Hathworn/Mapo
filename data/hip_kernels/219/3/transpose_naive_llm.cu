#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_naive(float *odata, float *idata, int width, int height)
{
    // Utilize shared memory to optimize memory access pattern
    __shared__ float tile[TILE_DIM][TILE_DIM+1];
    
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Load data from global memory to shared memory
    if (xIndex < width && yIndex < height)
    {
        unsigned int index_in = xIndex + width * yIndex;
        tile[threadIdx.y][threadIdx.x] = idata[index_in];
    }
    
    __syncthreads();
    
    // Calculate transpose indices
    xIndex = blockDim.y * blockIdx.y + threadIdx.x;
    yIndex = blockDim.x * blockIdx.x + threadIdx.y;
    
    // Write data from shared memory to global memory
    if (xIndex < height && yIndex < width)
    {
        unsigned int index_out = xIndex + height * yIndex;
        odata[index_out] = tile[threadIdx.x][threadIdx.y];
    }
}