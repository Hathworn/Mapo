#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_DIM 16

__global__ void transpose(float *odata, float *idata, int width, int height)
{
    __shared__ float block[BLOCK_DIM][BLOCK_DIM+1];

    // Optimize reading by initializing indices only once
    unsigned int xIndex = blockIdx.x * BLOCK_DIM + threadIdx.x;
    unsigned int yIndex = blockIdx.y * BLOCK_DIM + threadIdx.y;

    // Read the matrix tile into shared memory
    if ((xIndex < width) && (yIndex < height))
    {
        block[threadIdx.y][threadIdx.x] = idata[yIndex * width + xIndex];
    }

    __syncthreads();

    // Transpose and write the matrix tile to global memory
    xIndex = blockIdx.y * BLOCK_DIM + threadIdx.x;
    yIndex = blockIdx.x * BLOCK_DIM + threadIdx.y;

    if ((xIndex < height) && (yIndex < width))
    {
        odata[yIndex * height + xIndex] = block[threadIdx.x][threadIdx.y];
    }
}