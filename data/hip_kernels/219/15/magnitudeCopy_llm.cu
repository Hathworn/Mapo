#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void magnitudeCopy(float *mag_vec, float *vec, const int n) {
    // Use shared memory for faster access
    __shared__ float sharedVec[1024];
    
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Load data into shared memory
    if (xIndex < n) {
        sharedVec[threadIdx.x] = vec[xIndex];
        __syncthreads(); // Ensure all threads have loaded data
        
        // Calculate magnitude
        mag_vec[xIndex] = abs(sharedVec[threadIdx.x]);
    }
}