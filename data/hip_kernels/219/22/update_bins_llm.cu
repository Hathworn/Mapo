#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_bins(float *vec, int *bin, int *bin_counters, const int num_bins, const int n, const float slope, const float intercept)
{
    // Compute index and ensure it is within bounds
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (xIndex >= n) return;

    // Calculate the new bin value
    float temp = fabsf(vec[xIndex]); // Use fast absolute value function
    int bin_new_val = (temp > (intercept * 0.000001f)) ? slope * (intercept - temp) : num_bins;

    // Update bin and bin_counters only if there is a change
    if (bin[xIndex] != bin_new_val)
    {
        if (bin[xIndex] < num_bins)
            atomicSub(&bin_counters[bin[xIndex]], 1); // Use atomicSub for clarity

        if (bin_new_val < num_bins)
            atomicAdd(&bin_counters[bin_new_val], 1);

        bin[xIndex] = bin_new_val;
    }
}