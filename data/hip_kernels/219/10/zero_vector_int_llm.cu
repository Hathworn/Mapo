#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void zero_vector_int(int *vec, const int n)
{
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Use int4 to set four elements at a time
    int4* vec4 = reinterpret_cast<int4*>(vec);
    unsigned int xIndex4 = xIndex / 4;

    if (xIndex4 < n / 4) {
        int4 zeroVal = make_int4(0, 0, 0, 0);
        vec4[xIndex4] = zeroVal;
    }

    // Handle remaining elements if n is not a multiple of 4
    if (xIndex < n) {
        vec[xIndex] = 0;
    }
}