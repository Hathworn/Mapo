#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void int_copy(int *vec_to, const int *vec_from, const int n)
{
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory to optimize memory transfer
    extern __shared__ int shared_mem[];
    
    if (xIndex < n) {
        // Load data into shared memory
        shared_mem[threadIdx.x] = vec_from[xIndex];
        __syncthreads();

        // Store data from shared memory to global memory
        vec_to[xIndex] = shared_mem[threadIdx.x];
    }
}