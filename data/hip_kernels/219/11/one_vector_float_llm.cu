#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void one_vector_float(float *vec, const int n)
{
    // Use a single instruction to calculate the index
    unsigned int xIndex = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Check if index is within bounds and assign value using a conditional operator
    if (xIndex < n) { vec[xIndex] = 1.0f; } 
}