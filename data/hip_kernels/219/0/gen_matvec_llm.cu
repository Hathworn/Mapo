#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gen_matvec(float *A, float *x, float *y, const int m, const int n)
{
    // Use shared memory for input vector
    extern __shared__ float shared_x[];
    
    // Calculate global index
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (threadIdx.x < n) {
        // Load data to shared memory
        shared_x[threadIdx.x] = x[threadIdx.x];
    }
    
    __syncthreads(); // Synchronize threads
    
    if (xIndex < m) {
        float c = 0.0f;
        for (int i = 0; i < n; i++) {
            // Accessing shared memory
            c += shared_x[i] * A[xIndex + m * i];
        }
        y[xIndex] = c;
    }
}