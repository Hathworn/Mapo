#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void segCountSum(int *counter, int *segcounter, const int countlength)
{
    unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Use grid-stride loop for better utilization of GPU resources
    for (int idx = xIndex; idx < countlength; idx += blockDim.x * gridDim.x)
    {
        int sum = 0;
        for (int jj = 0; jj < countlength; jj++)
        {
            sum += segcounter[idx + jj * countlength];
        }
        counter[idx] = sum;
    }
}