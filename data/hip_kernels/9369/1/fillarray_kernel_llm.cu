#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fillarray_kernel(float *x, float v, int np) {
    // Calculate a unique thread identifier for the 1D grid
    int ii = threadIdx.x + blockIdx.x * blockDim.x;

    // Use grid-stride loop for better memory coalescing
    for (; ii < np; ii += blockDim.x * gridDim.x) {
        x[ii] = v;
    }
}