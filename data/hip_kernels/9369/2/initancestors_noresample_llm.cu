#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initancestors_noresample(int *ancestor, int np) {
    int ii = threadIdx.x + blockIdx.x * blockDim.x;  // Correct thread index calculation using blockDim
    int stride = blockDim.x * gridDim.x;             // Compute stride for the grid
    for(; ii < np; ii += stride) {                   // Use a for-loop with calculated stride
        ancestor[ii] = ii;                           // Assign current index to ancestor array
    }
}