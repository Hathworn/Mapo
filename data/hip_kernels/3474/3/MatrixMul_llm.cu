#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMul(float *darray_1, float *darray_2 , float *dres_arr, int n) {
    // Calculate row and column indices
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure the thread is within matrix boundaries
    if (row < n && col < n) {
        float sum = 0.0f; // Use a local variable for accumulation
        for (int i = 0; i < n; i++) {
            sum += darray_1[row * n + i] * darray_2[i * n + col];
        }
        dres_arr[row * n + col] = sum; // Write the result to global memory
    }
}