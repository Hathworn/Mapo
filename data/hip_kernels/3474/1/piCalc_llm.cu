#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void piCalc(double *area, double width, int rects) {
    // Use registers instead of variables for improved performance
    double mid, height;
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Unroll the loop to reduce overhead
    for (; index < rects; index += blockDim.x * gridDim.x) {
        mid = (index + 0.5) * width; // Calculate midpoint for current index
        height = 4.0 / (1.0 + mid * mid); // Calculate height using original pi algorithm
        area[index] += height; // Accumulate the height to area
    }
}