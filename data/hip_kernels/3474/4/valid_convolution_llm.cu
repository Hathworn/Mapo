#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void convolution(int conv_col, int conv_row, float *d_kernel, int k_size, float *d_matrix, int size_x, int size_y, float *d_conv, int max_row, int max_col) {
    int conv_index = conv_col + conv_row * max_col;
    float sum = 0; // Use a local variable to reduce global memory access
    for (int k_row = 0; k_row < k_size; k_row++) {
        for (int k_col = 0; k_col < k_size; k_col++) {
            sum += d_kernel[k_col + (k_row * k_size)] * d_matrix[(conv_col + k_col) + (conv_row + k_row) * size_x];
        }
    }
    d_conv[conv_index] = sum;
}

__global__ void valid_convolution(float *d_kernel, int k_size, float *d_matrix, int size_x, int size_y, float *d_conv, int max_row, int max_col) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (row < max_row && col < max_col) { // Reorder checks to optimize for short-circuit evaluation
        convolution(col, row, d_kernel, k_size, d_matrix, size_x, size_y, d_conv, max_row, max_col);
    }
}