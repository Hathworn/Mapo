#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(const int *a, const int *b, int *c){
    // Use shared memory to minimize global memory access
    __shared__ int sum_shared;

    // Calculate sum using shared memory
    sum_shared = *a + *b;

    // Store result back to global memory
    *c = sum_shared;
}