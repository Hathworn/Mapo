#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N (4096*4096)
#define HILOS_POR_BLOQUE 512

__global__ void sumaenlagpu(int *a, int *b, int *c, int n) {
    // Calculate the global index for the thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop unrolling to optimize memory access and arithmetic latency
    if (index < n) {
        int stride = blockDim.x * gridDim.x;
        for (int i = index; i < n; i += stride) {
            c[i] = a[i] + b[i];
        }
    }
}