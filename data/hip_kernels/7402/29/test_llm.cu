#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void test()
{
    // Use local variable to reduce shared memory contention
    unsigned int localCounter = atomicInc(d_PointCounter, 0x7fffffff);
    
    // Perform computation with localCounter
    printf("cnt : %d \n", localCounter);
}