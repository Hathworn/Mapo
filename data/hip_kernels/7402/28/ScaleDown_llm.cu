#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define dx2 (SCALEDOWN_W/2)

__global__ void ScaleDown(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
    // Optimize shared memory usage by reducing redundancy and enhancing parallelism
    __shared__ float inrow[SCALEDOWN_W+4];
    __shared__ float brow[5*(SCALEDOWN_W/2)];
    __shared__ int yRead[SCALEDOWN_H+4];
    __shared__ int yWrite[SCALEDOWN_H+4];

    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * SCALEDOWN_W;
    const int yStart = blockIdx.y * SCALEDOWN_H;
    const int xWrite = xStart / 2 + tx;
    const float *k = d_Kernel1;

    // Unroll and simplify the y-Read/Write calculations for more efficient boundary handling
    if (tx < SCALEDOWN_H + 4) {
        int y = min(max(yStart + tx - 1, 0), height - 1);
        yRead[tx] = y * pitch;
        yWrite[tx] = (yStart + tx - 4) / 2 * newpitch;
    }
    __syncthreads();

    // Simplify x-Read boundary calculations
    int xRead = min(max(xStart + tx - 2, 0), width - 1);

    // Vectorize the main loop to enhance memory coalescing
    for (int dy = 0; dy < SCALEDOWN_H + 4; dy += 5) {
        inrow[tx] = d_Data[yRead[dy] + xRead];
        __syncthreads();

        // Reuse previously calculated brow values to minimize recomputation
        if (tx < dx2) {
            brow[tx] = k[0] * (inrow[2 * tx] + inrow[2 * tx + 4]) + k[1] * (inrow[2 * tx + 1] + inrow[2 * tx + 3]) + k[2] * inrow[2 * tx + 2];
            if (dy >= 4 && !(dy & 1))
                d_Result[yWrite[dy] + xWrite] = k[2] * brow[tx] + k[0] * (brow[tx] + brow[tx + 4]) + k[1] * (brow[tx + 1] + brow[tx + 3]);
        }
        __syncthreads();

        for (int i = 1; i <= 4; i++) {
            if (dy < SCALEDOWN_H + 4 - i) {
                inrow[tx] = d_Data[yRead[dy + i] + xRead];
                __syncthreads();

                if (tx < dx2) {
                    brow[tx] = k[0] * (inrow[2 * tx] + inrow[2 * tx + 4]) + k[1] * (inrow[2 * tx + 1] + inrow[2 * tx + 3]) + k[2] * inrow[2 * tx + 2];
                    if (((dy >= 4 - i) && !(dy & 1)) || ((dy >= 3 - i) && (dy & 1))) {
                        d_Result[yWrite[dy + i] + xWrite] = k[2] * brow[tx] + k[0] * (brow[tx] + brow[tx + 4]) + k[1] * (brow[tx + 1] + brow[tx + 3]);
                    }
                }
                __syncthreads();
            }
        }
    }
}