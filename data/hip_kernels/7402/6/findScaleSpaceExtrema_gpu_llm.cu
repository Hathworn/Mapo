#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void addpoint() {}

__device__ unsigned int d_PointCounter[1];
__device__ float *pd[60];
__device__ float *pgpyr[72];
__device__ int temsize;
__constant__ int d_oIndex[36];

static const int BLOCK_SIZE_ONE_DIM = 32;
static const int Descript_BLOCK_SIZE = 32;

__global__ void test_gpu(int pitch, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y < height)
    {
        pgpyr[0][y*pitch + x] += 100;
    }
    if (y*pitch + x < 5)
    {
        printf("%f\n", pd[0][y*pitch + x]);
    }
}

__global__ void findScaleSpaceExtrema_gpu(float *d_point, int p_pitch, int s, int width, int pitch, int height, const int threshold, const int nOctaveLayers, const int maxNum)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Border checks
    if (y >= height - SIFT_IMG_BORDER || x >= width - SIFT_IMG_BORDER || x < SIFT_IMG_BORDER || y < SIFT_IMG_BORDER)
        return;

    float *currptr = pd[s] + y * pitch + x;
    float *prevptr = pd[s - 1] + y * pitch + x;
    float *nextptr = pd[s + 1] + y * pitch + x;

    float val = *currptr;
    int step = pitch;

    // Neighborhood comparison for extrema
    if (std::abs(val) > threshold &&
        ((val > 0 && \
        val >= max(currptr[-1], currptr[1], currptr[-step-1], currptr[-step], currptr[-step+1], currptr[step-1], currptr[step], currptr[step+1], \
        nextptr[0], nextptr[-1], nextptr[1], nextptr[-step-1], nextptr[-step], nextptr[-step+1], nextptr[step-1], nextptr[step], nextptr[step+1], \
        prevptr[0], prevptr[-1], prevptr[1], prevptr[-step-1], prevptr[-step], prevptr[-step+1], prevptr[step-1], prevptr[step], prevptr[step+1])) || \
        (val < 0 && \
        val <= min(currptr[-1], currptr[1], currptr[-step-1], currptr[-step], currptr[-step+1], currptr[step-1], currptr[step], currptr[step+1], \
        nextptr[0], nextptr[-1], nextptr[1], nextptr[-step-1], nextptr[-step], nextptr[-step+1], nextptr[step-1], nextptr[step], nextptr[step+1], \
        prevptr[0], prevptr[-1], prevptr[1], prevptr[-step-1], prevptr[-step], prevptr[-step+1], prevptr[step-1], prevptr[step], prevptr[step+1]))))
    {
        // Use shared memory for repeated parameters
        const float img_scale = 1.f / (255 * SIFT_FIXPT_SCALE);
        const float deriv_scale = img_scale * 0.5f;
        const float second_deriv_scale = img_scale;
        const float cross_deriv_scale = img_scale * 0.25f;

        int j = 0, layer;
        float dx, dy, ds, dxx, dyy, dxy;
        float Vs = 0, Vx = 0, Vy = 0, contr = 0;

        for (; j < SIFT_MAX_INTERP_STEPS; j++)
        {
            currptr = pd[s] + y * pitch + x;
            prevptr = pd[s - 1] + y * pitch + x;
            nextptr = pd[s + 1] + y * pitch + x;

            // Calculate derivatives
            dx = (currptr[1] - currptr[-1]) * deriv_scale;
            dy = (currptr[pitch] - currptr[-pitch]) * deriv_scale;
            ds = (nextptr[0] - prevptr[0]) * deriv_scale;
            float v2 = currptr[0] * 2;

            dxx = (currptr[1] + currptr[-1] - v2) * second_deriv_scale;
            dyy = (currptr[pitch] + currptr[-pitch] - v2) * second_deriv_scale;
            float dss = (nextptr[0] + prevptr[0] - v2) * second_deriv_scale;
            dxy = (currptr[pitch+1] - currptr[1-pitch] - currptr[-1+pitch] + currptr[-pitch-1]) * cross_deriv_scale;

            float dxs = (nextptr[1] - nextptr[-1] - prevptr[1] + prevptr[-1]) * cross_deriv_scale;
            float dys = (nextptr[pitch] - nextptr[-pitch] - prevptr[pitch] + prevptr[-pitch]) * cross_deriv_scale;

            // Precompute some values
            float idxx = dyy * dss - dys * dys;
            float idxy = dys * dxs - dxy * dss;
            float idxs = dxy * dys - dyy * dxs;
            float idet = __fdividef(1.0f, idxx * dxx + idxy * dxy + idxs * dxs);
            float idyy = dxx * dss - dxs * dxs;
            float idys = dxy * dxs - dxx * dys;
            float idss = dxx * dyy - dxy * dxy;

            // Calculate offsets
            float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
            float pdy = idet * (idxy * dx + idyy * dy + idys * ds);
            float pds = idet * (idxs * dx + idys * dy + idss * ds);

            Vx = -pdx; Vy = -pdy; Vs = -pds;

            // Break condition if offsets are within 0.5
            if (std::abs(Vs) < 0.5f && std::abs(Vx) < 0.5f && std::abs(Vy) < 0.5f)
                break;

            // Update positions
            x += int(Vx > 0 ? (Vx + 0.5) : (Vx - 0.5));
            y += int(Vy > 0 ? (Vy + 0.5) : (Vy - 0.5));
            s += int(Vs > 0 ? (Vs + 0.5) : (Vs - 0.5));

            layer = s - o * (nOctaveLayers + 2);
            if (layer < 1 || layer > nOctaveLayers || y < SIFT_IMG_BORDER || y >= height - SIFT_IMG_BORDER || x < SIFT_IMG_BORDER || x >= width - SIFT_IMG_BORDER)
                return;
        }
        if (j >= SIFT_MAX_INTERP_STEPS)
            return;

        // Contrast and curvature checks
        float t = dx * Vx + dy * Vy + ds * Vs;
        contr = currptr[0] * img_scale + t * 0.5f;
        if (std::abs(contr) * nOctaveLayers < 0.04)
            return;

        float tr = dxx + dyy;
        float det = dxx * dyy - dxy * dxy;
        if (det <= 0 || tr * tr * 10 >= (10 + 1) * (10 + 1) * det)
            return;

        // Store keypoints
        unsigned int idx = atomicInc(d_PointCounter, 0x7fffffff);
        idx = (idx >= maxNum ? maxNum - 1 : idx);

        d_point[idx] = (x + Vx) * (1 << o);
        d_point[idx + p_pitch * 1] = (y + Vy) * (1 << o);
        d_point[idx + p_pitch * 2] = o + (layer<<8) + ((int)(((Vs + 0.5) * 255) + 0.5) << 16);
        float size = 1.6 * __powf(2.f, (layer + Vs) / nOctaveLayers) * (1 << o) * 2;
        d_point[idx + p_pitch * 3] = size;
        d_point[idx + p_pitch * 4] = std::abs(contr);
    }
}