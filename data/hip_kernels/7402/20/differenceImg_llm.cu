#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void differenceImg(float *d_Octave0, float *d_Octave1, float *d_diffOctave, int pitch, int height) {
    // Calculate pixel index only if it is within bounds
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Use conditional to limit global memory access
    if (y < height && x < pitch) {
        int index = y * pitch + x;
        d_diffOctave[index] = d_Octave1[index] - d_Octave0[index];
    }
}