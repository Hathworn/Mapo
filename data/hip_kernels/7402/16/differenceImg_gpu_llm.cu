#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void differenceImg_gpu(const float* img1, const float* img2, float* result, int numElements)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x; // Calculate global thread index

    if (index < numElements) { // Boundary check
        result[index] = img1[index] - img2[index]; // Compute difference for each pixel
    }
}