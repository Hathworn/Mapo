#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void permute_colors_kernel(int num_rows, int *row_colors, int *color_permutation)
{
    // Calculate global thread index
    int row_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a simpler loop increment to enhance performance
    while (row_id < num_rows)
    {
        int color = row_colors[row_id];
        // Use cached memory access if available
        #if __CUDA_ARCH__ >= 350
        color = __ldg(color_permutation + color);
        #else
        color = color_permutation[color];
        #endif
        row_colors[row_id] = color;

        // Move to the next row processed by this thread
        row_id += blockDim.x * gridDim.x;
    }
}