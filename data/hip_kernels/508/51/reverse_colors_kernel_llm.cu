#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reverse_colors_kernel(int num_rows, int max_color, int *row_colors)
{
    int row_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop for better performance on large arrays
    int stride = blockDim.x * gridDim.x;
    for ( ; row_id < num_rows ; row_id += stride ) 
    {
        int color = row_colors[row_id];

        if (color > 0)
        {
            // Reverse color: 1 -> max_color, max_color -> 1
            color = max_color - color + 1;
        }
        
        row_colors[row_id] = color;
    }
}