#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void toGrayScale(unsigned char *output, const unsigned char *input, int width, int height, int components)
{
    // Calculate the global thread coordinates
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Return if out of image bounds
    if (row >= height || column >= width)
        return;

    // Compute the 1D index for the image data
    int index = (column + row * width) * components;  // Modify index calculation for better readability
    unsigned char greyScale = __fmaf_rn(0.114f, input[index], __fmaf_rn(0.587f, input[index + 1], 0.299f * input[index + 2]));

    // Store the computed gray scale value
    output[column + row * width] = greyScale;
}