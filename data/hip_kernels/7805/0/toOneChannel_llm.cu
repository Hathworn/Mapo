#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void toOneChannel(unsigned char *data, int width, int height, int components)
{
    // Calculate flattened index for the current thread
    int index = blockIdx.y * blockDim.y * width + blockIdx.x * blockDim.x + threadIdx.y * width + threadIdx.x;

    // Calculate column and row based on the index
    int column = index % width;
    int row = index / width;

    // Return if the current position is out of bounds
    if (row >= height || column >= width)
        return;

    unsigned char *threadData = data + (components * index);

    for (int i = 0; i < components - 1; i++)
    {
        threadData[i] = 0;
    }
}