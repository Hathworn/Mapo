#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void toGrayScale(unsigned char *output, unsigned char *input, int width, int height, int components)
{
    // Cache the block index calculations.
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < height && column < width)  // Combined bounds check for early return.
    {
        int index = column + row * width;
        unsigned char *threadData = input + components * index;
        
        // Reorganize reads, compute greyScale directly.
        unsigned char greyScale = 0.114f * threadData[0] + 0.587f * threadData[1] + 0.299f * threadData[2];

        output[index] = greyScale;  // Direct access using index, eliminating additional pointer arithmetic.
    }
}