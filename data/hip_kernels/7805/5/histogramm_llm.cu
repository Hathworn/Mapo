#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogramm(float* hist, unsigned char* input, int width, int height, int stride)
{
    int index = blockIdx.x * blockDim.x * stride + threadIdx.x;
    int size = width * height;

    // Exit early if index is out of range
    if (index >= size)
        return;

    __shared__ unsigned int histo_private[256];

    // Initialize shared memory histogram to 0
    #pragma unroll
    for (int i = 0; i < 8; i++)
    {
        histo_private[threadIdx.x * 8 + i] = 0;
    }

    __syncthreads();

    // Process input and update the shared histogram
    while (index < size)
    {
        int pixel = input[index];
        atomicAdd(&(histo_private[pixel]), 1);
        index += blockDim.x * stride; // Correct stride calculation
    }

    __syncthreads();

    // Update global histogram using shared histogram
    #pragma unroll
    for (int i = 0; i < 8; i++)
    {
        int x_off = threadIdx.x * 8 + i;
        hist[x_off * 3 + 0] = (x_off - 128.f) / 256.f * (float)width;

        float factor = .48f;
        float scaledValue = ((float)(histo_private[x_off]) / (float)size) - (factor / gridDim.x);
        atomicAdd(&(hist[x_off * 3 + 1]), scaledValue * (float)height);
    }
}