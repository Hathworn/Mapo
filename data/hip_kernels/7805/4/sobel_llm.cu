#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sobel(unsigned char *output, unsigned char *input, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y >= height || x >= width)
        return;

    const int BLOCK_SIZE = 16;

    // Shared Data with boundary padding
    __shared__ unsigned char pixels[BLOCK_SIZE + 2][BLOCK_SIZE + 2];

    // Linear index for loading into shared memory
    int threadIndex = threadIdx.x + threadIdx.y * blockDim.x;
    int loadX = threadIndex % (BLOCK_SIZE + 2);
    int loadY = threadIndex / (BLOCK_SIZE + 2);

    // Load data into shared memory
    if (loadY < BLOCK_SIZE + 2 && loadX < BLOCK_SIZE + 2) {
        int globalX = blockIdx.x * blockDim.x + loadX - 1;
        int globalY = blockIdx.y * blockDim.y + loadY - 1;

        // Clamp global indices
        globalX = max(0, min(globalX, width - 1));
        globalY = max(0, min(globalY, height - 1));

        pixels[loadY][loadX] = input[globalY * width + globalX];
    }
    __syncthreads();

    // Sobel weights
    float weightsX[3][3] = { {-1, -2, -1}, {0, 0, 0}, {1, 2, 1} };
    float weightsY[3][3] = { {-1, 0, 1}, {-2, 0, 2}, {-1, 0, 1} };

    float pointX = 0.f;
    float pointY = 0.f;

    // Compute Sobel filter
    #pragma unroll
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            int indexX = threadIdx.x + 1 + j;
            int indexY = threadIdx.y + 1 + i;
            unsigned char pixel = pixels[indexY][indexX];
            pointX += pixel * weightsX[i][j];
            pointY += pixel * weightsY[i][j];
        }
    }

    // Write output
    int index = x + y * width;
    if (x < width && y < height) {
        output[index] = sqrtf(pointX * pointX + pointY * pointY);
    }
}