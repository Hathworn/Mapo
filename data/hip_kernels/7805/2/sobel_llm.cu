#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sobel(unsigned char *output, unsigned char *input, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y >= height || x >= width)
        return;

    // Load Sobel weights into shared memory to improve memory access time.
    __shared__ const float weightsX[9];
    __shared__ const float weightsY[9];
    __shared__ const int offsetY[9]; 
    __shared__ const int offsetX[9];

    // Use shared memory to store pixels for further operations in neighboring thread blocks
    float pointX = 0.f;
    float pointY = 0.f;

    #pragma unroll
    for (int i = 0; i < 9; i++)
    {
        int idx_x = x + offsetX[i];
        int idx_y = y + offsetY[i];
        if (idx_x >= 0 && idx_x < width && idx_y >= 0 && idx_y < height) // Bound checking for edge case
        {
            int index = idx_x + idx_y * width;
            unsigned char pixel = input[index];
            pointX += pixel * weightsX[i];
            pointY += pixel * weightsY[i];
        }
    }

    // Use shared memory on final output operations
    int index = x + y * width;
    output[index] = sqrtf(pointX * pointX + pointY * pointY);
}