#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define domain_width    4096
#define domain_height   2048

__global__ void stencil_kernel(float *x_new, float *x_old) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim for block size
    int y = blockIdx.y * blockDim.y + threadIdx.y; // Use blockDim for block size

    // Avoid recomputation by calculating index once
    int idx = y * domain_width + x;
    
    if (y > 0 && y < domain_height - 1 && x > 0 && x < domain_width - 1) {
        float sum = x_old[idx] + 
                    x_old[idx - 1] +
                    x_old[idx + 1] +
                    x_old[idx + domain_width] +
                    x_old[idx - domain_width];

        x_new[idx] = sum / 5.0f; // Calculate new value
    }
}