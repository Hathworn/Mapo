#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmul_kernel(float *C, float *A, float *B) {

    extern __shared__ float shared[];
    float* sA = &shared[0];
    float* sB = &shared[block_size_y * tile_size_y * block_size_x];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockIdx.x * block_size_x * tile_size_x + tx;
    int y = blockIdx.y * block_size_y * tile_size_y + ty;
    int k, kb;

    float sum[tile_size_y][tile_size_x] = {0}; // Initialize sums

    for (k = 0; k < WIDTH; k += block_size_x) {

        __syncthreads();

        #pragma unroll
        for (int i = 0; i < tile_size_y; i++) {
            // Load tiles into shared memory
            sA[(ty + block_size_y * i) * block_size_x + tx] = A[(y + i * block_size_y) * WIDTH + k + tx];

            #pragma unroll
            for (int j = 0; j < tile_size_x; j++) {
                sB[(ty + block_size_y * i) * (block_size_x * tile_size_x) + (tx + j * block_size_x)] = 
                    B[(k + ty + block_size_y * i) * WIDTH + (x + j * block_size_x)];
            }
        }

        __syncthreads();

        #pragma unroll
        for (kb = 0; kb < block_size_x; kb++) {
            #pragma unroll
            for (int i = 0; i < tile_size_y; i++) {
                #pragma unroll
                for (int j = 0; j < tile_size_x; j++) {
                    // Compute local product
                    sum[i][j] += sA[(ty + block_size_y * i) * block_size_x + kb] * 
                                 sB[kb * (block_size_x * tile_size_x) + (tx + j * block_size_x)];
                }
            }
        }
    }

    // Store result
    #pragma unroll
    for (int i = 0; i < tile_size_y; i++) {
        #pragma unroll
        for (int j = 0; j < tile_size_x; j++) {
            C[(y + block_size_y * i) * WIDTH + (x + j * block_size_x)] = sum[i][j];
        }
    }
}