#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmul_kernel(float *C, float *A, float *B) {

    __shared__ float sA[block_size_y * tile_size_y][block_size_x];
    __shared__ float sB[block_size_y * tile_size_y][block_size_x * tile_size_x];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockIdx.x * block_size_x * tile_size_x + threadIdx.x;
    int y = blockIdx.y * block_size_y * tile_size_y + threadIdx.y;

    float sum[tile_size_y][tile_size_x] = {0}; // Initialize in declaration

    for (int k = 0; k < WIDTH; k += block_size_x) {

        __syncthreads();
        #pragma unroll
        for (int i = 0; i < tile_size_y; i++) {
            int row_a = y + i * block_size_y;
            int row_b = k + ty + block_size_y * i; 

            sA[ty + block_size_y * i][tx] = A[row_a * WIDTH + k + tx];

            #pragma unroll
            for (int j = 0; j < tile_size_x; j++) {
                sB[ty + block_size_y * i][tx + j * block_size_x] = B[row_b * WIDTH + x + j * block_size_x];
            }
        }
        __syncthreads();

        // Compute
        #pragma unroll
        for (int kb = 0; kb < block_size_x; kb++) {
            #pragma unroll
            for (int i = 0; i < tile_size_y; i++) {
                float valA = sA[ty + block_size_y * i][kb]; // Cache value from sA
                #pragma unroll
                for (int j = 0; j < tile_size_x; j++) {
                    sum[i][j] += valA * sB[kb][tx + j * block_size_x];
                }
            }
        }
    }

    // Store result
    #pragma unroll
    for (int i = 0; i < tile_size_y; i++) {
        #pragma unroll
        for (int j = 0; j < tile_size_x; j++) {
            C[y * WIDTH + x + block_size_y * i * WIDTH + j * block_size_x] = sum[i][j];
        }
    }
}