#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cn_pnpoly_naive(int* bitmap, float2* points, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        int c = 0;
        float2 p = points[i];
        
        // Unroll loop through vertices to reduce branching
        float2 vj = d_vertices[VERTICES-1];
        for (int j = 0; j < VERTICES; j++) {
            float2 vk = d_vertices[j];
            if ((vj.y > p.y) != (vk.y > p.y)) { // Simplify vertical check
                float slope = (vk.x - vj.x) / (vk.y - vj.y);
                if (p.x < slope * (p.y - vj.y) + vj.x) {
                    c = !c;
                }
            }
            vj = vk; // Move to next vertex
        }

        bitmap[i] = c; // Set bitmap value
    }
}