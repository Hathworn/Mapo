#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmul_kernel(float *C, float *A, float *B) {
    __shared__ float sA[block_size_y][block_size_x];
    __shared__ float sB[block_size_y][block_size_x];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockIdx.x * block_size_x + tx;
    int y = blockIdx.y * block_size_y + ty;

    float sum = 0.0;
    int k;

    // Load data into shared memory in a coalesced manner
    for (k = 0; k < WIDTH; k += block_size_x) {
        sA[ty][tx] = A[y * WIDTH + k + tx];
        sB[ty][tx] = B[(k + ty) * WIDTH + x];
        __syncthreads();

        // Unrolling loop for better performance
        #pragma unroll
        for (int kb = 0; kb < block_size_x; kb++) {
            sum += sA[ty][kb] * sB[kb][tx];
        }
        __syncthreads();
    }

    C[y * WIDTH + x] = sum;
}