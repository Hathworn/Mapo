#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define TILE_WIDTH 16

__global__ void matmul_kernel(float *C, float *A, float *B) {
    __shared__ float shared_A[TILE_WIDTH][TILE_WIDTH]; // Shared memory for A
    __shared__ float shared_B[TILE_WIDTH][TILE_WIDTH]; // Shared memory for B

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int x = blockIdx.x * TILE_WIDTH + tx;
    int y = blockIdx.y * TILE_WIDTH + ty;

    float sum = 0.0;

    for (int m = 0; m < WIDTH / TILE_WIDTH; ++m) {
        // Load tiles into shared memory
        if (y < WIDTH && (m * TILE_WIDTH + tx) < WIDTH)
            shared_A[ty][tx] = A[y * WIDTH + m * TILE_WIDTH + tx];
        else
            shared_A[ty][tx] = 0.0;

        if (x < WIDTH && (m * TILE_WIDTH + ty) < WIDTH)
            shared_B[ty][tx] = B[(m * TILE_WIDTH + ty) * WIDTH + x];
        else
            shared_B[ty][tx] = 0.0;

        __syncthreads(); // Wait for both tiles to be loaded

        // Multiply the two tiles
        for (int k = 0; k < TILE_WIDTH; ++k) {
            sum += shared_A[ty][k] * shared_B[k][tx];
        }

        __syncthreads(); // Ensure all threads are done using current tiles
    }

    if (y < WIDTH && x < WIDTH) // Ensure C index is within bounds
        C[y * WIDTH + x] = sum;
}