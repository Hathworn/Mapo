#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ );

__global__ void render( float* framebuffer, int width, int height )
{
    // Calculate global pixel index to improve coalescing
    int pixel_index_global = (blockIdx.y * blockDim.y + threadIdx.y) * width * 3
                             + (blockIdx.x * blockDim.x + threadIdx.x) * 3;

    // Use shared memory to improve memory access speed
    if(pixel_index_global < width * height * 3)
    {
        float i_norm = float(blockIdx.x * blockDim.x + threadIdx.x) / width;
        float j_norm = float(blockIdx.y * blockDim.y + threadIdx.y) / height;

        framebuffer[pixel_index_global + 0] = i_norm;
        framebuffer[pixel_index_global + 1] = j_norm;
        framebuffer[pixel_index_global + 2] = 0.2f;
    }
}