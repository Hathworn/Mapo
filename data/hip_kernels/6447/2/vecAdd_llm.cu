#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(float* __restrict__ C, const float* __restrict__ A, const float* __restrict__ B, int n) {
    // Use shared memory for better efficiency
    extern __shared__ float sharedMemory[];

    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n) {
        // Load data into shared memory
        sharedMemory[tid] = A[id];
        __syncthreads(); // Synchronize to ensure all data is loaded

        // Perform vector addition
        C[id] = sharedMemory[tid] + B[id];
    }
}