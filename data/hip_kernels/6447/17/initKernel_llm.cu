#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void initKernel(double* data, int count, double val) {
    int ti = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for loop unrolling

    // Use loop unrolling for better memory access and coalescing
    for (int idx = ti; idx < count; idx += stride) {
        data[idx] = val;
    }
}