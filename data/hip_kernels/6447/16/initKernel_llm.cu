#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initKernel(double* data, int count, double val) {
    // Use block-level shared memory for better bandwidth utilization (if applicable)
    int ti = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll the loop if possible
    #pragma unroll
    for (int idx = ti; idx < count; idx += blockDim.x * gridDim.x) {
        data[idx] = val;
    }
}