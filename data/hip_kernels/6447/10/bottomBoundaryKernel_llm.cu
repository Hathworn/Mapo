#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bottomBoundaryKernel(double* temperature, int block_size) {
    // Calculate global index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Precompute constant stride
    int stride = block_size + 2;
    
    // Check if within boundary limits
    if (i < block_size) {
        // Set temperature value at the boundary
        temperature[stride * (block_size + 1) + (1 + i)] = 1.0;
    }
}