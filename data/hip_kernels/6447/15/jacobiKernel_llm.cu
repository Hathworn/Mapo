#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void jacobiKernel(double* temperature, double* new_temperature, int block_size) {
    // Calculate the global thread indices
    int i = (blockDim.x * blockIdx.x + threadIdx.x) + 1;
    int j = (blockDim.y * blockIdx.y + threadIdx.y) + 1;
    
    // Check bounds and perform the computation
    if (i <= block_size && j <= block_size) {
        int index = j * (block_size + 2) + i; // Precomputing to avoid redundant calculations
        int left = index - 1;
        int right = index + 1;
        int up = index - (block_size + 2);
        int down = index + (block_size + 2);

        // Calculate new temperature value
        new_temperature[index] = (
            temperature[left] +
            temperature[right] +
            temperature[up] +
            temperature[down] +
            temperature[index]
        ) * DIVIDEBY5; // Use of constant to avoid magic numbers
    }
}