#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize the global kernel function
__global__ void leftPackingKernel(double* temperature, double* ghost, int block_size) {
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure all threads that write into ghost array are within bounds
    if (j < block_size) {
        int idx = (block_size + 2) * (1 + j) + 1;  // Precompute repeated index calculation
        ghost[j] = temperature[idx];
    }
}