#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void leftUnpackingKernel(double* temperature, double* ghost, int block_size) {
    // Calculate the global id for the thread
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    // Use if condition inside the loop to ensure all threads hit the condition check
    if (j < block_size) {
        // Load ghost value into the temperature array
        temperature[(block_size + 2) * (1 + j) + 1] = ghost[j];
    }
}