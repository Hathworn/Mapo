#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil2DKernel(double* temperature, double* new_temperature, int block_x, int block_y, int thread_size) {
    int i_start = (blockDim.x * blockIdx.x + threadIdx.x) * thread_size + 1;
    int i_finish = (blockDim.x * blockIdx.x + threadIdx.x) * thread_size + thread_size;
    int j_start = (blockDim.y * blockIdx.y + threadIdx.y) * thread_size + 1;
    int j_finish = (blockDim.y * blockIdx.y + threadIdx.y) * thread_size + thread_size;

    // Use shared memory for performance optimization
    __shared__ double shared_temperature[32][32]; // Adjust TILE_SIZE according to hardware limits

    for (int i = i_start; i <= i_finish; i++) {
        for (int j = j_start; j <= j_finish; j++) {
            if (i <= block_x && j <= block_y) {
                // Load data into shared memory
                shared_temperature[threadIdx.x][threadIdx.y] = temperature[j * (block_x + 2) + i];
                __syncthreads();

                // Compute new values using shared memory to reduce global memory access
                new_temperature[j * (block_x + 2) + i] = (
                    ((threadIdx.x > 0) ? shared_temperature[threadIdx.x - 1][threadIdx.y] : temperature[j * (block_x + 2) + (i - 1)]) +
                    ((threadIdx.x < blockDim.x - 1) ? shared_temperature[threadIdx.x + 1][threadIdx.y] : temperature[j * (block_x + 2) + (i + 1)]) +
                    ((threadIdx.y > 0) ? shared_temperature[threadIdx.x][threadIdx.y - 1] : temperature[(j - 1) * (block_x + 2) + i]) +
                    ((threadIdx.y < blockDim.y - 1) ? shared_temperature[threadIdx.x][threadIdx.y + 1] : temperature[(j + 1) * (block_x + 2) + i]) +
                    shared_temperature[threadIdx.x][threadIdx.y]
                ) * DIVIDEBY5;

                __syncthreads();
            }
        }
    }
}