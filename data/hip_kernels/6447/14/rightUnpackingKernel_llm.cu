#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rightUnpackingKernel(double* temperature, double* ghost, int block_size) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory when possible to reduce repetitive global memory access
    __shared__ double local_ghost[256]; // assuming max blockDim.x is 256

    if (j < block_size) {
        // Load data from global memory to shared memory
        local_ghost[threadIdx.x] = ghost[j];
    }

    __syncthreads(); // Ensure all threads have loaded their values

    if (j < block_size) {
        // Perform the unpacking using shared memory
        temperature[(block_size + 2) * (1 + j) + block_size] = local_ghost[threadIdx.x];
    }
}