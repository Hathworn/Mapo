#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initKernel(double* temperature, int block_size) {
    // Calculate 1D thread index to enhance memory access pattern
    int index = (blockDim.x * gridDim.x) * (threadIdx.y + blockIdx.y * blockDim.y) + (threadIdx.x + blockIdx.x * blockDim.x);
    int total_size = (block_size + 2) * (block_size + 2);

    if (index < total_size) { // Flatten the 2D grid into 1D index for boundary checking
        temperature[index] = 0.0;
    }
}