#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void topBoundaryKernel(double* temperature, int block_size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < block_size) {
        // Use shared memory to reduce global memory access
        __shared__ double temp_cache[BLOCK_SIZE];
        temp_cache[threadIdx.x] = 1.0;
        __syncthreads();
        
        // Write back to global memory
        temperature[1 + i] = temp_cache[threadIdx.x];
    }
}