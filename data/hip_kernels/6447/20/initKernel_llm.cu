#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initKernel(double* data, int count, double val) {
    // Cache commonly used indices
    int ti = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll loop for performance
    for (int i = ti; i < count; i += stride) {
        data[i] = val;
    }
}