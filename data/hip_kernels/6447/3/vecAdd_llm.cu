#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(float* C, float* A, float* B, int n) {
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread is within bounds and perform vector addition
    if (id < n) {
        C[id] = A[id] + B[id];
    }
}