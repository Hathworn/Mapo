#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rightPackingKernel(double* temperature, double* ghost, int block_size) {
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory for improved access speed within block
    extern __shared__ double shared_temp[];

    // Load data into shared memory if within bounds
    if (j < block_size) {
        shared_temp[threadIdx.x] = temperature[(block_size + 2) * (1 + j) + block_size];
    }

    // Ensure all threads have loaded their data
    __syncthreads();

    // Write data from shared memory to ghost array
    if (j < block_size) {
        ghost[j] = shared_temp[threadIdx.x];
    }
}