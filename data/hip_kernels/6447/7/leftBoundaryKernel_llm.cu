#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void leftBoundaryKernel(double* temperature, int block_size) {
    // Use threadIdx.x directly, no calculation needed for j
    if (threadIdx.x < block_size) {
        // Calculate base index outside loop to prevent recalculation
        int baseIndex = (block_size + 2);
        // Eliminate multiplication inside the loop
        temperature[baseIndex + baseIndex * threadIdx.x] = 1.0;
    }
}