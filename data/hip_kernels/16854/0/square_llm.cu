#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(int *array, int n) {
    // Use block index and thread index for better parallelism
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n)
        array[tid] = array[tid] * array[tid];
}