#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float adaptive_mapping(float k, float q, float val_pixel){
    return (k * log(1 + val_pixel)) / ((100 * log10(1 + maxLum)) * (powf((log(2 + 8 * (val_pixel / maxLum))), (log(q) / log(0.5)))));
}

__global__ void tonemap_adaptive(float* imageIn, float* imageOut, int width, int height, int channels, int depth, float q, float k) {
    // Optimize by calculating the pixel index once
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if (Row < height && Col < width) {
        int index = (Row * width + Col) * 3;
        // Avoid redundant image index calculations
        imageOut[index + BLUE] = adaptive_mapping(k, q, imageIn[index + BLUE]);
        imageOut[index + GREEN] = adaptive_mapping(k, q, imageIn[index + GREEN]);
        imageOut[index + RED] = adaptive_mapping(k, q, imageIn[index + RED]);
    }
}