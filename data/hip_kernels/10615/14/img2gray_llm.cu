#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput) {
    // Calculate row and col once
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single if condition for bounds check
    if ((row < height) && (col < width)) {
        int idx = row * width + col;
        int rgb_idx = idx * 3;

        // Improved memory access coalescing and reduced computation redundancy
        unsigned char r = imageInput[rgb_idx + RED];
        unsigned char g = imageInput[rgb_idx + GREEN];
        unsigned char b = imageInput[rgb_idx + BLUE];

        imageOutput[idx] = r * 0.299f + g * 0.587f + b * 0.114f;
    }
}