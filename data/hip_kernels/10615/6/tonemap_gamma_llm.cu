#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float gamma_correction(float f_stop, float gamma, float val)
{
    return powf((val * powf(2, f_stop)), (1.0 / gamma));
}
__global__ void tonemap_gamma(float* imageIn, float* imageOut, int width, int height, int channels, int depth, float f_stop, float gamma)
{
    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    int Col = blockDim.x * blockIdx.x + threadIdx.x;

    if(Row < height && Col < width) {
        int idx = (Row * width + Col) * 3; // Calculate base index once
        // Apply gamma correction efficiently
        imageOut[idx + BLUE] = gamma_correction(f_stop, gamma, imageIn[idx + BLUE]);
        imageOut[idx + GREEN] = gamma_correction(f_stop, gamma, imageIn[idx + GREEN]);
        imageOut[idx + RED] = gamma_correction(f_stop, gamma, imageIn[idx + RED]);
    }
}