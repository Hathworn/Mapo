#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_maximum_kernel(float *array, int *mutex, unsigned int n, int blockSize) {
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int stride = gridDim.x * blockDim.x;
    unsigned int offset = 0;

    extern __shared__ float cache[];

    float temp = -1.0;
    while (index + offset < n) {
        temp = fmaxf(temp, array[index + offset]);
        offset += stride;
    }

    cache[threadIdx.x] = temp;
    __syncthreads();

    // Optimized reduction using warp reduction technique
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            cache[threadIdx.x] = fmaxf(cache[threadIdx.x], cache[threadIdx.x + s]);
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        while (atomicCAS(mutex, 0, 1) != 0);  // lock
        maxLum = fmaxf(maxLum, cache[0]);
        atomicExch(mutex, 0);  // unlock
    }
}