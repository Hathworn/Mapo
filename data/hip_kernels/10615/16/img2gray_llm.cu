#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define RED 0
#define GREEN 1
#define BLUE 2

__global__ void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput) {
    // Calculate pixel position using thread and block indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to load pixel data for faster access
    __shared__ unsigned char sharedImage[1024]; // Adjust size based on block dimensions

    int imgIndex = (row * width + col) * 3;

    // Load pixel data into shared memory
    if ((row < height) && (col < width)) {
        sharedImage[threadIdx.y * blockDim.x + threadIdx.x] = imageInput[imgIndex + RED] * 0.299
                                                               + imageInput[imgIndex + GREEN] * 0.587
                                                               + imageInput[imgIndex + BLUE] * 0.114;
        __syncthreads();

        // Write grayscale value to output
        imageOutput[row * width + col] = sharedImage[threadIdx.y * blockDim.x + threadIdx.x];
    }
}