#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define RED 0
#define GREEN 1
#define BLUE 2

__global__ void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate the index only once
    int idx = row * width + col;

    if (row < height && col < width) {
        // Use memory coalescing by accessing imageInput with a calculated offset
        unsigned char r = imageInput[idx * 3 + RED];
        unsigned char g = imageInput[idx * 3 + GREEN];
        unsigned char b = imageInput[idx * 3 + BLUE];

        // Direct writing to imageOutput
        imageOutput[idx] = static_cast<unsigned char>(r * 0.299f + g * 0.587f + b * 0.114f);
    }
}