#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define RED 0
#define GREEN 1
#define BLUE 2

__global__ void PictureKernell(unsigned char *imageInput, int width, int height, unsigned char *imageOutput) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within image bounds
    if (row < height && col < width) {
        int idx = (row * width + col) * 3;  // Calculate 1D index for 3-channel image
        // Grayscale conversion with fixed point arithmetic
        imageOutput[row * width + col] = imageInput[idx + RED] * 0.299f + 
                                         imageInput[idx + GREEN] * 0.587f + 
                                         imageInput[idx + BLUE] * 0.114f;
    }
}
```
