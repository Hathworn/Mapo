#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized for better readability and performance
__global__ void PictureKernell(unsigned char *d_Pin, unsigned char *d_Pout, int n, int m) {
    // Calculate row and column index for each thread
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use single index to avoid recalculating Row*n
    int index = Row * n + Col;

    // Ensure threads are within image bounds
    if (Row < m && Col < n) {
        d_Pout[index] = 2 * d_Pin[index]; // Perform operation and write result
    }
}