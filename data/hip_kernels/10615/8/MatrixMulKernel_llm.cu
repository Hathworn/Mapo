#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16  // Assuming TILE_WIDTH is pre-defined

__global__ void MatrixMulKernel(float *d_M, float *d_N, float *d_P, int width) {
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x; 
    int by = blockIdx.y;
    int tx = threadIdx.x; 
    int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    float Pvalue = 0.0f;

    // Unroll the loop over width/TILE_WIDTH
    for (int i = 0; i < width / TILE_WIDTH; ++i) {
        // Load elements into shared memory
        if (row < width && (i * TILE_WIDTH + tx) < width)
            Mds[ty][tx] = d_M[row * width + i * TILE_WIDTH + tx];
        else
            Mds[ty][tx] = 0.0f;
        
        if ((i * TILE_WIDTH + ty) < width && col < width)
            Nds[ty][tx] = d_N[(i * TILE_WIDTH + ty) * width + col];
        else
            Nds[ty][tx] = 0.0f;
            
        __syncthreads();

        // Compute partial product
        #pragma unroll
        for (int j = 0; j < TILE_WIDTH; ++j) {
            Pvalue += Mds[ty][j] * Nds[j][tx];
        }
        __syncthreads();
    }

    // Write the result back to the global memory
    if (row < width && col < width)
        d_P[row * width + col] = Pvalue;
}