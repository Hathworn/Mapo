#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define RED 0
#define GREEN 1
#define BLUE 2

__global__ void img2gray(unsigned char *imageInput, int width, int height, unsigned char *imageOutput) {
    // Use shared memory for performance improvement
    extern __shared__ unsigned char sharedMem[];
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * width + col;
    
    if (row < height && col < width) {
        // Load pixel data into shared memory
        sharedMem[threadIdx.y * blockDim.x * 3 + threadIdx.x * 3 + RED] = imageInput[idx * 3 + RED];
        sharedMem[threadIdx.y * blockDim.x * 3 + threadIdx.x * 3 + GREEN] = imageInput[idx * 3 + GREEN];
        sharedMem[threadIdx.y * blockDim.x * 3 + threadIdx.x * 3 + BLUE] = imageInput[idx * 3 + BLUE];
        __syncthreads();

        // Calculate gray value from shared memory
        unsigned char red = sharedMem[threadIdx.y * blockDim.x * 3 + threadIdx.x * 3 + RED];
        unsigned char green = sharedMem[threadIdx.y * blockDim.x * 3 + threadIdx.x * 3 + GREEN];
        unsigned char blue = sharedMem[threadIdx.y * blockDim.x * 3 + threadIdx.x * 3 + BLUE];
        
        imageOutput[idx] = red * 0.299f + green * 0.587f + blue * 0.114f;
    }
}