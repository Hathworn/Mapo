#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure i is within bounds and process threads that fall within numElements
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}