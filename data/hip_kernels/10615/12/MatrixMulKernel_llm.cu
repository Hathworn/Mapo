#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulKernel(float *d_M, float *d_N, float *d_P, int width) {
    // Allocate shared memory for input matrices
    __shared__ float M_tile[16][16];
    __shared__ float N_tile[16][16];

    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    float Pvalue = 0;

    // Loop over tiles
    for (int tile = 0; tile < (width + 15) / 16; ++tile) {
        // Load data into shared memory
        if (Row < width && tile * 16 + threadIdx.x < width) {
            M_tile[threadIdx.y][threadIdx.x] = d_M[Row * width + tile * 16 + threadIdx.x];
        } else {
            M_tile[threadIdx.y][threadIdx.x] = 0.0;
        }

        if (Col < width && tile * 16 + threadIdx.y < width) {
            N_tile[threadIdx.y][threadIdx.x] = d_N[(tile * 16 + threadIdx.y) * width + Col];
        } else {
            N_tile[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads();

        // Compute partial product for the tile
        for (int i = 0; i < 16; ++i) {
            Pvalue += M_tile[threadIdx.y][i] * N_tile[i][threadIdx.x];
        }

        __syncthreads();
    }
    
    // Write back the result
    if (Row < width && Col < width) {
        d_P[Row * width + Col] = Pvalue;
    }
}