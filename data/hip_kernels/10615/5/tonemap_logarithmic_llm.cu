#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float logarithmic_mapping(float k, float q, float val_pixel) {
    return (log10(1 + q * val_pixel)) / (log10(1 + k * maxLum));
}

__global__ void tonemap_logarithmic(float* imageIn, float* imageOut, int width, int height, int channels, int depth, float q, float k) {
    // Calculate one-dimensional index for better memory coalescing
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensuring idx is within bounds of the image
    if (idx < width * height) {
        int pixelOffset = idx * 3;  // Each pixel has 3 channels: BLUE, GREEN, RED
        imageOut[pixelOffset + BLUE] = logarithmic_mapping(k, q, imageIn[pixelOffset + BLUE]);
        imageOut[pixelOffset + GREEN] = logarithmic_mapping(k, q, imageIn[pixelOffset + GREEN]);
        imageOut[pixelOffset + RED] = logarithmic_mapping(k, q, imageIn[pixelOffset + RED]);
    }
}