#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(int *dest, int a, int b) {
    // Use thread and block indices for parallel execution
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid == 0) { // Ensure only one thread performs the addition
        *dest = a + b;
    }
}