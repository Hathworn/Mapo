#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vector_add(double const *A_dev, double const *B_dev, double *C_dev, int const N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Ensure the range is checked before accessing the arrays
    if (i < N) {
        C_dev[i] = A_dev[i] + B_dev[i];
    }
}