#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtract_kernel(float *in1, float *in2, float *out)
{
    // Compute the linear global thread ID
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Linearize the thread ID across the three dimensions
    long int id = (k * c_Size.y + j) * c_Size.x + i;
    
    // Check if the thread ID is within the valid range
    if (i < c_Size.x && j < c_Size.y && k < c_Size.z)
    {
        // Perform the subtraction operation
        out[id] = in1[id] - in2[id];
    }
}