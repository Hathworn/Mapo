#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_normalize_and_add_to_output(float *dev_vol_in, float *dev_vol_out, float *dev_accumulate_weights, float *dev_accumulate_values)
{
    // Calculate linear thread index based on a flattened 3D grid
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Early exit for out-of-bounds threads in the grid
    if (i >= c_volSize.x || j >= c_volSize.y || k >= c_volSize.z)
    {
        return;
    }

    // Compute linear index in a row-major 3D structure
    long int out_idx = i + (j + k * c_volSize.y) * c_volSize.x;

    float eps = 1e-6;

    // Conditional normalization based on the flag and weight magnitude
    if (c_normalize)
    {
        float weight = dev_accumulate_weights[out_idx];
        if (abs(weight) > eps)
        {
            dev_vol_out[out_idx] = dev_vol_in[out_idx] + (dev_accumulate_values[out_idx] / weight);
        }
        else
        {
            dev_vol_out[out_idx] = dev_vol_in[out_idx];
        }
    }
    else
    {
        dev_vol_out[out_idx] = dev_vol_in[out_idx] + dev_accumulate_values[out_idx];
    }
}