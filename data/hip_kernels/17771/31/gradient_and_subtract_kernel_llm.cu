#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void gradient_and_subtract_kernel(float *in, float *grad_x, float *grad_y, float *grad_z)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Check if the indices are within bounds
    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
        return;

    // Calculate linear index
    long int id = (k * c_Size.y + j) * c_Size.x + i;

    // Prefetching adjacent indices for x, y, z directions
    long int id_x = id + 1;
    long int id_y = id + c_Size.x;
    long int id_z = id + c_Size.x * c_Size.y;

    // Compute gradients and subtract only if within bounds
    if (i < c_Size.x - 1)
        grad_x[id] -= (in[id_x] - in[id]) / c_Spacing.x;
    if (j < c_Size.y - 1)
        grad_y[id] -= (in[id_y] - in[id]) / c_Spacing.y;
    if (k < c_Size.z - 1)
        grad_z[id] -= (in[id_z] - in[id]) / c_Spacing.z;
}