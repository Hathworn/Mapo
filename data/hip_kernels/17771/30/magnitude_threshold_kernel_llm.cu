#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void magnitude_threshold_kernel(float * grad_x, float * grad_y, float * grad_z, float gamma)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Ensure thread only operates within valid bounds
    if (i < c_Size.x && j < c_Size.y && k < c_Size.z) 
    {
        long int id = (k * c_Size.y + j) * c_Size.x + i;

        // Compute magnitude using more efficient intrinsic functions
        float norm = sqrtf(grad_x[id] * grad_x[id] + grad_y[id] * grad_y[id] + grad_z[id] * grad_z[id]);
        
        // Perform threshold operation based on norm
        if (norm > gamma)
        {
            float ratio = gamma / norm;
            grad_x[id] *= ratio;
            grad_y[id] *= ratio;
            grad_z[id] *= ratio;
        }
    }
}