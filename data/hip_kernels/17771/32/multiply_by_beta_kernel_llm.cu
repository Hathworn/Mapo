#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiply_by_beta_kernel(float *input, float *output, float beta)
{
    // Combine thread and block ids into a single linear index for clearer range checks
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idz = blockIdx.z * blockDim.z + threadIdx.z;

    // Use a flattened index directly, reduce computations
    unsigned long int id = (idz * c_Size.y + idy) * c_Size.x + idx;

    // Optimize boundary check logic
    if (idx < c_Size.x && idy < c_Size.y && idz < c_Size.z)
    {
        output[id] = input[id] * beta;
    }
}