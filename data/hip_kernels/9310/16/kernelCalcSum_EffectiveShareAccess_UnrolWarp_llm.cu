#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void kernelCalcSum_EffectiveShareAccess_UnrolWarp(const int* dataArray, int arraySize, int* sum)
{
    __shared__ int cache[1024]; // Pre-allocate shared memory with a fixed size

    int cacheIndex = threadIdx.x;
    int arrayIndex1 = blockDim.x * blockIdx.x + threadIdx.x;
    int arrayIndex2 = arrayIndex1 + gridDim.x * blockDim.x;

    // Initialize cache using a single write per thread
    int tempSum = 0;
    if (arrayIndex1 < arraySize)
    {
        tempSum += dataArray[arrayIndex1];
    }

    if (arrayIndex2 < arraySize)
    {
        tempSum += dataArray[arrayIndex2];
    }
    cache[cacheIndex] = tempSum;

    __syncthreads();

    int blockSize = blockDim.x;
    for (int offset = blockSize >> 1; offset > 32; offset >>= 1)
    {
        if (cacheIndex < offset)
        {
            cache[cacheIndex] += cache[cacheIndex + offset];
        }
        __syncthreads();
    }
    
    // Unroll the final warp for performance
    if (cacheIndex < 32)
    {
        volatile int* vcache = cache;  // Use volatile to prevent optimizer reordering
        vcache[cacheIndex] += vcache[cacheIndex + 32];
        vcache[cacheIndex] += vcache[cacheIndex + 16];
        vcache[cacheIndex] += vcache[cacheIndex + 8];
        vcache[cacheIndex] += vcache[cacheIndex + 4];
        vcache[cacheIndex] += vcache[cacheIndex + 2];
        vcache[cacheIndex] += vcache[cacheIndex + 1];
    }

    if (cacheIndex == 0)
    {
        atomicAdd(sum, cache[0]);
    }
}