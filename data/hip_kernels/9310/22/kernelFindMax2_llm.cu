#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void kernelFindMax2(const int* dataArray, int arraySize, int* maxVal)
{
    __shared__ int cache[1024];  // Optimize shared memory allocation size

    int cacheIndex = threadIdx.x;
    int arrayIndex = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Load elements into cache with bounds checking
    int temp = (arrayIndex < arraySize) ? dataArray[arrayIndex] : INT_MIN;
    cache[cacheIndex] = temp;
    __syncthreads();

    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) 
    {
        if (cacheIndex < offset) 
        {
            cache[cacheIndex] = max(cache[cacheIndex], cache[cacheIndex + offset]);
        }
        __syncthreads();
    }

    // Only one thread updates the global maximum
    if (cacheIndex == 0)
    {
        atomicMax(maxVal, cache[0]);
    }
}