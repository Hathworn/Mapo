#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void kernelCalcSum_ShareMem(const int* dataArray, int arraySize, int* sum)
{
    __shared__ int cache[1024]; // Use fixed shared memory size for known dimensions.

    int cacheIndex = threadIdx.x;
    int arrayIndex = blockDim.x * blockIdx.x + threadIdx.x;

    cache[cacheIndex] = (arrayIndex < arraySize) ? dataArray[arrayIndex] : 0; // Simplify conditional store.
    __syncthreads();

    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) // Reduce in shared memory.
    {
        if (cacheIndex < offset)
        {
            cache[cacheIndex] += cache[cacheIndex + offset];
        }
        __syncthreads();
    }

    if (cacheIndex == 0)
    {
        atomicAdd(sum, cache[0]); // Efficient atomic addition for final sum.
    }
}