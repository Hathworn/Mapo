#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void kernelFindMax5(const int* dataArray, int arraySize, int* maxVal)
{
    // Use dynamic shared memory allocation
    extern __shared__ int cache[];

    int cacheIndex = threadIdx.x;

    // Calculate two array indices for the thread
    int arrayIndex1 = blockDim.x * blockIdx.x + threadIdx.x;
    int arrayIndex2 = arrayIndex1 + gridDim.x * blockDim.x;

    // Initialize shared memory to minimum integer value
    int threadMax = INT_MIN;

    // Load data into shared memory
    if (arrayIndex1 < arraySize)
    {
        threadMax = max(threadMax , dataArray[arrayIndex1]);
    }

    if (arrayIndex2 < arraySize)
    {
        threadMax = max(threadMax , dataArray[arrayIndex2]);
    }
    
    // Store the maximum found by this thread
    cache[cacheIndex] = threadMax;
    
    __syncthreads();

    // Parallel reduction to find block's max
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1)
    {
        if (cacheIndex < offset)
        {
            cache[cacheIndex] = max(cache[cacheIndex], cache[cacheIndex + offset]);
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (cacheIndex == 0)
    {
        atomicMax(maxVal, cache[0]);
    }
}