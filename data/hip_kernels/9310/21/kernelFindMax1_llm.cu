#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void kernelFindMax1(const int* dataArray, int arraySize, int* maxVal)
{
    // Use shared memory for intermediate max values
    extern __shared__ int sharedMax[];

    int threadIndex = threadIdx.x;
    int blockIndex = blockIdx.x;
    int blockSize = blockDim.x;
    int arrayIndex = blockSize * blockIndex + threadIndex;

    // Each thread initializes shared memory
    int localMax = (arrayIndex < arraySize) ? dataArray[arrayIndex] : INT_MIN;
    sharedMax[threadIndex] = localMax;
    __syncthreads();

    // Perform reduction to find block maximum
    for (int stride = blockSize / 2; stride > 0; stride /= 2) {
        if (threadIndex < stride) {
            sharedMax[threadIndex] = max(sharedMax[threadIndex], sharedMax[threadIndex + stride]);
        }
        __syncthreads();
    }

    // Use atomic operation to update global maximum from block maximums
    if (threadIndex == 0) {
        atomicMax(maxVal, sharedMax[0]);
    }
}