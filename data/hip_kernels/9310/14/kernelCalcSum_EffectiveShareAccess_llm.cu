#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void kernelCalcSum_EffectiveShareAccess(const int* dataArray, int arraySize, int* sum)
{
    __shared__ int cache[1024]; // Predefined shared memory size for optimization

    int cacheIndex = threadIdx.x;

    int arrayIndex = blockDim.x * blockIdx.x + threadIdx.x;
    int value = (arrayIndex < arraySize) ? dataArray[arrayIndex] : 0;

    cache[cacheIndex] = value;

    __syncthreads();

    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if (cacheIndex < offset)
        {
            cache[cacheIndex] += cache[cacheIndex + offset]; // Fixed addressing for coalesced access
        }
        __syncthreads();
    }

    if (cacheIndex == 0)
    {
        atomicAdd(sum, cache[0]);
    }
}