#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void kernelFindMax4(const int* dataArray, int arraySize, int* maxVal)
{
    __shared__ extern int cache[];

    int cacheIndex = threadIdx.x;
    int arrayIndex1 = blockDim.x * blockIdx.x + threadIdx.x;
    int arrayIndex2 = arrayIndex1 + gridDim.x * blockDim.x;

    // Initialize shared memory to minimum integer value
    cache[cacheIndex] = INT_MIN;

    // Load data from global memory and compute max into shared memory
    if (arrayIndex1 < arraySize) {
        cache[cacheIndex] = max(cache[cacheIndex], dataArray[arrayIndex1]);
    }
    if (arrayIndex2 < arraySize) {
        cache[cacheIndex] = max(cache[cacheIndex], dataArray[arrayIndex2]);
    }

    __syncthreads();

    // Intra-block reduction to find maximum within block
    for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
        if (cacheIndex < offset) {
            cache[cacheIndex] = max(cache[cacheIndex], cache[cacheIndex + offset]);
        }
        __syncthreads();
    }

    // Atomic operation to update global maximum
    if (cacheIndex == 0) {
        atomicMax(maxVal, cache[0]);
    }
}