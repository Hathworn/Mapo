#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void kernelFindMax3(const int* dataArray, int arraySize, int* maxVal)
{
    extern __shared__ int cache[];

    int cacheIndex = threadIdx.x;
    int arrayIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Load data into shared memory with boundary check
    cache[cacheIndex] = (arrayIndex < arraySize) ? dataArray[arrayIndex] : INT_MIN;

    __syncthreads();

    // Reduction to find max using warp-wise reduction
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) 
    {
        if (cacheIndex < offset)
        {
            cache[cacheIndex] = max(cache[cacheIndex], cache[cacheIndex ^ offset]);
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (cacheIndex == 0)
    {
        atomicMax(maxVal, cache[0]);
    }
}