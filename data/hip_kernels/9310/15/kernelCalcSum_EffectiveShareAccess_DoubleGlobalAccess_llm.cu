#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void kernelCalcSum_EffectiveShareAccess_DoubleGlobalAccess(const int* dataArray, int arraySize, int* sum)
{
    __shared__ int cache[1024]; // Pre-allocated shared memory to avoid dynamic allocation

    int cacheIndex = threadIdx.x;
    int arrayIndex = blockDim.x * blockIdx.x * 2 + threadIdx.x; // Optimize array access for coalesced memory access

    int tempSum = 0; // Use a register for temporary storage

    if (arrayIndex < arraySize)
    {
        tempSum += dataArray[arrayIndex];
    }

    int secondIndex = arrayIndex + blockDim.x;
    if (secondIndex < arraySize)
    {
        tempSum += dataArray[secondIndex];
    }

    cache[cacheIndex] = tempSum; // Store the sum in shared memory

    __syncthreads();

    int blockSize = blockDim.x;
    for (int offset = blockSize >> 1; offset > 0; offset >>= 1)
    {
        if (cacheIndex < offset)
        {
            cache[cacheIndex] += cache[cacheIndex + offset]; // Use index + offset for better readability
        }
        __syncthreads();
    }

    if (cacheIndex == 0)
    {
        atomicAdd(sum, cache[0]); // Atomic add the result of this block to the global sum
    }
}