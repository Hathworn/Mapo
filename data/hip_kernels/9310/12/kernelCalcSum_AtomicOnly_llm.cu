#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void kernelCalcSum_AtomicOnly(const int* __restrict__ dataArray, int arraySize, int* sum)
{
    // Use shared memory to reduce atomic operations on global memory
    extern __shared__ int sharedData[];
    
    int localIndex = threadIdx.x;
    int globalIndex = blockDim.x * blockIdx.x + localIndex;

    // Load data into shared memory
    sharedData[localIndex] = (globalIndex < arraySize) ? dataArray[globalIndex] : 0;
    __syncthreads();

    // Parallel reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (localIndex < stride) {
            sharedData[localIndex] += sharedData[localIndex + stride];
        }
        __syncthreads();
    }

    // Atomic add the result of each block to the global sum
    if (localIndex == 0) {
        atomicAdd(sum, sharedData[0]);
    }
}