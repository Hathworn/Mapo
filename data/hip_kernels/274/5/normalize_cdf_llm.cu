#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    // Cache normalization constant to improve memory access speed
    const float normalization_constant = 1.f / d_input_cdf[n - 1];
    
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if thread index is within bounds
    if (global_index_1d < n)
    {
        unsigned int input_value = d_input_cdf[global_index_1d];
        
        // Directly calculate and store normalized value
        d_output_cdf[global_index_1d] = input_value * normalization_constant;
    }
}