#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tonemap(float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new, float min_log_Y, float max_log_Y, float log_Y_range, int num_bins, int num_pixels_y, int num_pixels_x)
{
    int nx = num_pixels_x;
    int ny = num_pixels_y;
    int image_index = blockIdx.y * blockDim.y + threadIdx.y;
    int pixel_index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within image bounds
    if (pixel_index < nx && image_index < ny)
    {
        int image_index_1d = pixel_index + image_index * nx;
        float x = d_x[image_index_1d];
        float y = d_y[image_index_1d];
        float log_Y = d_log_Y[image_index_1d];
        
        // Calculate bin index and clamp within range
        int bin_index = min(num_bins - 1, int(num_bins * (log_Y - min_log_Y) / log_Y_range));
        float Y_new = d_cdf_norm[bin_index];

        // Calculate new RGB values
        float factor = Y_new / y;
        float X_new = x * factor;
        float Z_new = (1 - x - y) * factor;
        
        float r_new = X_new * 3.2406f + Y_new * -1.5372f + Z_new * -0.4986f;
        float g_new = X_new * -0.9689f + Y_new * 1.8758f + Z_new * 0.0415f;
        float b_new = X_new * 0.0557f + Y_new * -0.2040f + Z_new * 1.0570f;

        // Store the new RGB values
        d_r_new[image_index_1d] = r_new;
        d_g_new[image_index_1d] = g_new;
        d_b_new[image_index_1d] = b_new;
    }
}