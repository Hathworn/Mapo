#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_shared(float * v_new, const float * v) {
    extern __shared__ float s[];
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x + 1;
    
    // Improve memory coalescing by reading from global memory to shared memory efficiently
    s[tid] = v[id];
    
    // Only the first thread in the block handles boundary values
    if (threadIdx.x == 0) {
        int start = blockDim.x * blockIdx.x;
        int end = blockDim.x * gridDim.x;
        s[0] = (start > 0) ? v[start - 1] : v[start];
        int right = start + blockDim.x;
        s[blockDim.x + 1] = (right < end) ? v[right] : v[right - 1];
    }
    
    __syncthreads();
    
    // Perform the smoothing operation using shared memory
    v_new[id] = 0.25f * s[tid - 1] + 0.5f * s[tid] + 0.25f * s[tid + 1];
}