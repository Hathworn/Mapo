#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
    // Store predicate in shared memory
    int tid = threadIdx.x;
    s[tid] = p;
    __syncthreads();

    // Optimized reduction using warp shuffle instructions
    for (int offset = 16; offset > 0; offset /= 2) {
        p += __shfl_down_sync(0xFFFFFFFF, p, offset);
    }

    // Store result in shared memory only once
    if (tid == 0) {
        s[0] = p;
    }
    __syncthreads();
    
    return s[0]; // Return the sum from shared memory
}

__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    if (t == 0) {
        *d_out_shared = sr;
    }
}