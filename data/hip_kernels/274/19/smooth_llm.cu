#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth(float * v_new, const float * v) {
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x; // Correct block index calculation
    int numThreads = blockDim.x * gridDim.x;
    if (myIdx < numThreads) { // Check bounds to avoid illegal memory access
        int myLeftIdx = (myIdx == 0) ? 0 : myIdx - 1;
        int myRightIdx = (myIdx == (numThreads - 1)) ? numThreads - 1 : myIdx + 1;
        float myElt = v[myIdx];
        float myLeftElt = v[myLeftIdx];
        float myRightElt = v[myRightIdx];
        v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
    }
}