#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(float* d_out, float* d_in)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    float f = d_in[idx];
    d_out[idx] = f * f;
}