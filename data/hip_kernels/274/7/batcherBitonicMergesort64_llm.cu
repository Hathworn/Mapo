#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void swap(float& a, float& b)
{
    float temp = a;
    a = b;
    b = temp;
}
__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid  = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    for (int stage = 0; stage <= 5; stage++)
    {
        int div = 1 << (stage + 1); // Precompute division factor for indexing
        for (int substage = stage; substage >= 0; substage--)
        {
            int distance = 1 << substage; // Distance to value to be compared
            int comparison = tid - distance; // Value to be compared
            // Skip values not required for comparison; optimize by calculating division once
            if (comparison >= 0 && (comparison / div) == (tid / div)) {
                bool up = (comparison / div) % 2 == 1;
                // Use ternary operations for compact comparison and swapping
                if ((up && sdata[tid] > sdata[comparison]) || (!up && sdata[tid] < sdata[comparison])) {
                    swap(sdata[tid], sdata[comparison]);
                }
            }
        }
        __syncthreads();
    }

    d_out[tid] = sdata[tid];
}