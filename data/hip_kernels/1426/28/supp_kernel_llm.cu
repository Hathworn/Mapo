#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    int stride = gridDim.x * blockDim.x;  // Calculate grid stride
    for (int i = idx; i < N; i += stride) {  // Loop with grid stride
        if ((X[i * INCX] * X[i * INCX]) < (ALPHA * ALPHA)) {
            X[i * INCX] = 0;
        }
    }
}