#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[BLOCK];  // Define shared memory for thread block

    int id = threadIdx.x;
    int filter = blockIdx.x;

    local[id] = 0.0f;

    // Calculate the summation in parallel, unrolling the loop for efficiency
    for (int j = 0; j < batch; ++j) {
        int base_index = j * spatial * filters + filter * spatial + id;
        for (int i = 0; i < spatial; i += threads) {
            if (i + id < spatial) {
                local[id] += x[base_index + i];  // Accumulate values in shared memory
            }
        }
    }

    __syncthreads();

    // Reduce block-level summation using a single thread
    if (id == 0) {
        float sum = 0.0f;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        mean[filter] = sum / (spatial * batch);  // Compute mean
    }
}