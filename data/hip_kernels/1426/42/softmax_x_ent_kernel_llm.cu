#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Single calculation index for overall grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = blockDim.x * gridDim.x; // Stride calculation for grid-level looping

    // Loop through entire data set with grid-striding pattern
    for (; i < n; i += gridStride) {
        float t = truth[i];
        float p = pred[i];
        error[i] = t ? -log(p) : 0;
        delta[i] = t - p;
    }
}