#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x; 
    // Use single line execution to reduce branching instructions
    if(i < N) X[i * INCX] = ALPHA;
}