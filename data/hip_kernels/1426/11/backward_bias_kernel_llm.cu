#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define BLOCK size for shared memory
#define BLOCK 256

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Loop through batches with stride BLOCK to minimize memory latency
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index];  // Accumulate delta values
        }
    }
    part[p] = sum;

    __syncthreads();

    // Use atomicAdd to safely update bias_updates (avoid race condition)
    if (p == 0) {
        for (int i = 0; i < BLOCK; ++i) {
            atomicAdd(&bias_updates[filter], part[i]);
        }
    }
}