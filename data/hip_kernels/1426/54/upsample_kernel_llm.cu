#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    // Calculate the global thread index
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= N) return;

    // Precompute some values to avoid repeated calculations
    int w_stride = w * stride;
    int h_stride = h * stride;
    int wh = w * h;
    
    int out_w = index % w_stride;
    int out_h = (index / w_stride) % h_stride;
    int out_c = (index / (w_stride * h_stride)) % c;
    int b = index / (w_stride * h_stride * c);

    int in_w = out_w / stride;
    int in_h = out_h / stride;

    int in_index = b * wh * c + out_c * wh + in_h * w + in_w;

    // Optimized write or accumulate operations
    if(forward) {
        out[index] += scale * x[in_index];
    } else {
        atomicAdd(x + in_index, scale * out[index]);
    }
}