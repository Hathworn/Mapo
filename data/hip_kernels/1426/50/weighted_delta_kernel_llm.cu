#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function with conditional memory access
__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;

    // Process elements in a loop to handle more data per thread
    for (; i < n; i += totalThreads) {
        // Check bounds and perform operations
        if (i < n) {
            if (da) da[i] += dc[i] * s[i];
            if (db) db[i] += dc[i] * (1 - s[i]);
            ds[i] += dc[i] * (a[i] - b[i]);
        }
    }
}