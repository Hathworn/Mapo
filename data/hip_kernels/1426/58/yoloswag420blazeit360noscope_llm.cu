#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    // Calculate unique global index for each thread
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Check boundary condition 
    if (id < size) {
        // Apply conditional operation
        input[id] = (rand[id] < prob) ? 0 : input[id] * scale;
    }
}