#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Pre-calculate scale factor
    float scale = 1.f/(batch * spatial);

    // Optimize index calculation
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Use shared memory for partial sums (optional, depending on workload and memory constraints)
    extern __shared__ float shared_mean[];
    shared_mean[threadIdx.x] = 0;

    // Optimize loop order for better memory access and coalescing
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * spatial * filters + k * filters + i;
            shared_mean[threadIdx.x] += x[index];
        }
    }

    // Finalize the mean calculation
    mean[i] = shared_mean[threadIdx.x] * scale;
}