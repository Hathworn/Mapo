#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Optimize thread indexing for 1D grid
    if (i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);
        if (abs_val < 1.0f) {  // Use consistent floating-point literal
            error[i] = diff * diff;
            delta[i] = diff;
        } else {
            error[i] = 2.0f * abs_val - 1.0f;  // Use consistent floating-point literals
            delta[i] = copysignf(1.0f, diff);  // Optimize using copysignf for delta calculation
        }
    }
}