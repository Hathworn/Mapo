#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    int i = threadIdx.x;  // Use thread index for size dimension
    int j = blockIdx.x;   // Use block index for n dimension
    int k = blockIdx.y;   // Use grid dimension y for batch

    if (i < size && j < n && k < batch) {  // Check bounds
        output[(k * n + j) * size + i] += biases[j];  // Update output with bias
    }
}