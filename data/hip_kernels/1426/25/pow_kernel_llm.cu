#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize memory access by using loop unrolling
    int stride = gridDim.x * blockDim.x;
    for (; i < N; i += stride) {
        Y[i * INCY] = powf(X[i * INCX], ALPHA); // Use powf for performance
    }
}