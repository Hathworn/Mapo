#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Precompute constant values to reduce redundant calculations
    int wh = w * h;
    int out_c = c / (stride * stride);

    // Compute input indices
    int in_w = i % w;
    int in_h = (i / w) % h;
    int in_c = (i / wh) % c;
    int b = i / (wh * c);

    // Determine output indices based on the transformation
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Conditional assignment based on the direction
    if (forward)
        out[out_index] = x[in_index];
    else
        out[in_index] = x[out_index];
}