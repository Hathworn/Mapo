#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid-stride loop to handle more elements per thread
    for (; i < N; i += blockDim.x * gridDim.x)
    {
        X[i * INCX] = ALPHA;
    }
}