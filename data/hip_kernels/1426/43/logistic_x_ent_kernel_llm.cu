#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread is within bounds
    if(i < n) {
        float t = truth[i];
        float p = pred[i];
        
        // Compute error and delta
        error[i] = -t * logf(p + 1e-7f) - (1.0f - t) * logf(1.0f - p + 1e-7f);
        delta[i] = t - p;
    }
}