#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified thread index calculation
    if (i < N) {
        // Perform axpy operation if within bounds
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}