#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    float temp_sum = 0.0f; // Use a local variable to store reduction sum 

    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) { // Increment by threads directly
            int index = j * spatial * filters + filter * spatial + i;

            temp_sum += delta[index] * (x[index] - mean[filter]); // Accumulate values
        }
    }

    local[id] = temp_sum; // Store local sum
    __syncthreads();

    // Use parallel reduction to sum up all local[id] values
    for (int s = threads / 2; s > 0; s >>= 1) {
        if (id < s) {
            local[id] += local[id + s];
        }
        __syncthreads();
    }

    if (id == 0) {
        variance_delta[filter] = local[0] * -.5f * powf(variance[filter] + .00001f, -1.5f);
    }
}