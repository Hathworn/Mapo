#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate the global thread ID efficiently
    if (id >= size) return;

    int b = id / (minw * minh * minc);  // Calculate batch index
    int id_rem = id % (minw * minh * minc);
    int k = id_rem / (minw * minh);     // Calculate channel index
    id_rem %= (minw * minh);
    int j = id_rem / minw;              // Calculate height index
    int i = id_rem % minw;              // Calculate width index

    // Calculate the indexes in input/output arrays.
    int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));

    // Perform the weighted addition
    out[out_index] = s1 * out[out_index] + s2 * add[add_index];
}