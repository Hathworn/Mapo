#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation within a single dimension.
    int elements = (NX + NY) * B;  // Calculate total elements only once.
    if (i < elements) {
        int b = i / (NX + NY);
        int j = i % (NX + NY);
        if (j < NX) {
            if (X) X[b * NX + j] += OUT[i];  // Accumulate OUT to X.
        } else {
            if (Y) Y[b * NY + j - NX] += OUT[i];  // Accumulate OUT to Y.
        }
    }
}