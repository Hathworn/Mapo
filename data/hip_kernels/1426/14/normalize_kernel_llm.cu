#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for all threads

    for (int i = index; i < N; i += stride) // Use loop to handle more elements with fewer blocks
    {
        int f = (i / spatial) % filters;
        x[i] = (x[i] - mean[f]) / (sqrtf(variance[f] + 0.00001f));
    }
}