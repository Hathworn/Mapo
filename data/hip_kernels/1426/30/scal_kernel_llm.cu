#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use 1D grid for better memory access and simpler calculation
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N) X[i * INCX] *= ALPHA;
}