#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
  // Using shared memory for intermediate results
  extern __shared__ float shared[];
  float *shared_e = shared;  // For storing intermediate exponentiation results
  
  int i;
  float sum = 0.0f;
  float largest = -INFINITY;
  
  // Find largest element (reduce memory transactions)
  for(i = 0; i < n; ++i){
    float val = input[i*stride];
    largest = (val > largest) ? val : largest;
  }
  
  // Calculate softmax values
  for(i = 0; i < n; ++i){
    float e = expf(input[i*stride] / temp - largest / temp);
    sum += e;
    shared_e[i] = e; // Intermediate result stored in shared memory
  }
  
  // Normalize with the sum
  for(i = 0; i < n; ++i){
    output[i*stride] = shared_e[i] / sum;
  }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id >= batch * groups) return;

  int b = id / groups;
  int g = id % groups;

  // Launch device function with shared memory size set
  // Size of shared memory can be optimized based on 'n' and hardware capabilities
  softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}