#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation for 1D grid
    if (i < n) {
        float diff = truth[i] - pred[i];
        error[i] = fabsf(diff); // Use device-specific abs function
        delta[i] = copysignf(1.0f, diff); // Use copysignf to set delta
    }
}