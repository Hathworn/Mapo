#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    // Calculate the global index
    int s = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Return if index is out-of-bounds
    if (s >= size) return;

    float sum = 0.0f;
    
    // Optimize by unrolling loop
    #pragma unroll
    for (int i = 0; i < n; ++i) {
        // Accumulate absolute value
        sum += fabsf(input[i * size + s]);
    }
    
    // Compute mean 
    float mean = sum / n;
    
    // Optimize by using conditional operator only once
    float binarized_value = mean;
    if (input[s] <= 0) binarized_value = -mean;

    #pragma unroll
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = binarized_value;
    }
}