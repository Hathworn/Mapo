#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Process elements with a stride pattern to allow more threads to process
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        X[idx * INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[idx * INCX]));
    }
}