#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Use blockIdx.x only for 1D grid
    if (index >= n) return;

    float sum = 0.0f;
    // Unroll loop to improve performance
    for (int b = 0; b < batch; ++b) {
        int i = b * n + index;
        sum += delta[i];
    }

    bias_updates[index] += sum;
}