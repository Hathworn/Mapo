#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    int b = index / spatial;
    int i = index % spatial;

    float sum = 0;
    for (int f = 0; f < filters; ++f) {
        int linear_index = b * filters * spatial + f * spatial + i;
        sum += x[linear_index] * x[linear_index];  // Simplified sum calculation
    }
    
    sum = sqrtf(sum);
    sum = fmaxf(sum, 1.0f);  // Ensure sum is not zero

    for (int f = 0; f < filters; ++f) {
        int linear_index = b * filters * spatial + f * spatial + i;
        x[linear_index] /= sum;  // Normalize
        dx[linear_index] = (1.0f - x[linear_index]) / sum;  // Calculate dx
    }
}