#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Use shared memory for faster mean calculation
    __shared__ float sharedMean;
    float localMean = 0.0f;

    // Calculate mean in parallel using a single loop
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        localMean += fabsf(weights[f * size + i]);
    }
    atomicAdd(&sharedMean, localMean);

    __syncthreads();
    if (threadIdx.x == 0) {
        sharedMean /= size; // Compute average
    }
    __syncthreads();

    // Binarize weights using computed mean
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? sharedMean : -sharedMean;
    }
}