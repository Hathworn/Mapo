#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll the loop to process multiple elements per thread
    for(int index = i; index < N; index += blockDim.x * gridDim.x)
    {
        X[index * INCX] += ALPHA; // Modify only targeted elements
    }
}