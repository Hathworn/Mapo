#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculating global index for the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Looping to perform operations for multiple elements per thread if needed
    for(int index = i; index < n; index += blockDim.x * gridDim.x) {
        c[index] += a[index] * b[index];
    }
}