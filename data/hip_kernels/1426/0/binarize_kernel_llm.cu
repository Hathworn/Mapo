#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Use a 1D grid and 1D block for better thread allocation
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) { // Check boundary conditions to avoid overwriting
        binary[i] = (x[i] >= 0) ? 1.0f : -1.0f; // Ensure floating-point values
    }
}