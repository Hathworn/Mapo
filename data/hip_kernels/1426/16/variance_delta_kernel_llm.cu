#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Precompute constants for variance calculation
    float var_factor = -.5f * powf(variance[i] + .00001f, -1.5f);

    float sum = 0.0f;
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += delta[index] * (x[index] - mean[i]);
        }
    }
    
    // Multiply the sum by the precomputed variance factor
    variance_delta[i] = sum * var_factor;
}