#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    // Precompute common values outside the critical loop
    int f = (index / spatial) % filters;
    float inv_var_sqrt = 1.f / sqrtf(variance[f] + 0.00001f);
    float spatial_batch = spatial * batch;

    // Optimize redundant calculations in the kernel
    delta[index] = delta[index] * inv_var_sqrt 
                 + variance_delta[f] * 2.f * (x[index] - mean[f]) / spatial_batch 
                 + mean_delta[f] / spatial_batch;
}