#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = gridDim.x * blockDim.x;

    // Process all elements assigned to a thread using grid-stride loop
    for (; i < n; i += gridStride) {
        if (mask[i] == mask_num) {
            x[i] *= scale;
        }
    }
}