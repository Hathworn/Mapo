#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_THREADS_PER_BLOCK 256

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    float e;

    // Use shared memory to optimize access speed
    extern __shared__ float shared_mem[];
    float* shared_input = shared_mem;

    // Load input into shared memory
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        shared_input[i * stride] = input[i * stride];
    }
    __syncthreads();

    // Find the largest in shared memory for numerical stability
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        float val = shared_input[i * stride];
        atomicMax((int*)&largest, __float_as_int(val));
    }
    __syncthreads();

    // Compute exponentials and sum
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        e = expf(shared_input[i * stride] / temp - largest / temp);
        atomicAdd(&sum, e);
        output[i * stride] = e;
    }
    __syncthreads();

    // Normalize the output
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        output[i * stride] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;
    
    int s = id % spatial;
    id = id / spatial;
    int g = id % groups;
    int b = id / groups;
    int goff = group_offset[g] * spatial;
    int boff = b * stride;

    // Launch kernel with appropriate shared memory size
    int shared_mem_size = group_size[g] * sizeof(float);
    softmax_device<<<1, MAX_THREADS_PER_BLOCK, shared_mem_size>>>(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}