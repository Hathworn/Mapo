#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation, assuming 1D grid.
    if (index >= N) return;

    float B1_pow = powf(B1, t); // Precompute powf values outside loop.
    float B2_pow = powf(B2, t); 

    float mhat = m[index] / (1.f - B1_pow);
    float vhat = v[index] / (1.f - B2_pow);

    x[index] += rate * mhat / (sqrtf(vhat) + eps); // Simplify and use computation += syntax.
}