#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate unique thread index across grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;
    
    // Loop over elements in increments of grid size; leverage global memory coalescing
    for (; i < N; i += gridSize) {
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}