#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    // Calculate multi-dimensional indices
    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);
    
    int w_offset = -(size / 2);
    int h_offset = -(size / 2);

    int out_index = j + w * (i + h * (k + c * b));
    float delta_val = 0.0f; // Accumulate in a local variable

    // Unroll the loop for better performance
    #pragma unroll
    for(int l = 0; l < size; ++l) {
        int cur_h = h_offset + i + l;
        if(cur_h >= 0 && cur_h < h) { // Check height bounds once per row
            #pragma unroll
            for(int m = 0; m < size; ++m) {
                int cur_w = w_offset + j + m;
                if(cur_w >= 0 && cur_w < w) { // Check width bounds
                    int index = cur_w + w * (cur_h + h * (k + b * c));
                    delta_val += rate * (x[index] - x[out_index]);
                }
            }
        }
    }
    delta[out_index] = delta_val; // Assign accumulated value to global memory
}