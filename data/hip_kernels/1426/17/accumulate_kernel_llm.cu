#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Return if index is out of bounds for groups
    if (i >= groups) return;
    
    // Initialize sum for each group
    float temp_sum = 0.0f;
    
    // Unroll loop to optimize memory access and computation
    for (int k = 0; k < n; k++) {
        temp_sum += x[k * groups + i];
    }
    
    // Write result back
    sum[i] = temp_sum;
}
```
