#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Use shared memory for variance calculation
    extern __shared__ float shared_variance[];
    
    float scale = 1.f/(batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    
    // Initialize shared memory
    shared_variance[threadIdx.x] = 0;

    // Use striding over batch and spatial dimensions
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            shared_variance[threadIdx.x] += powf((x[index] - mean[i]), 2);
        }
    }
    
    // Write shared variance to global memory
    __syncthreads();
    variance[i] = shared_variance[threadIdx.x] * scale;
}