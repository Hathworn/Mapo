#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation

    // Loop to handle multiple elements per thread in case n is larger than grid size
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x) { 
        if (mask[idx] == mask_num) {
            x[idx] = val; // Update value if mask matches
        }
    }
}