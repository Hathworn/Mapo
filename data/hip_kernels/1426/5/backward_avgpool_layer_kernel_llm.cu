#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    // Calculate batch and channel from id
    int b = id / c;
    int k = id % c;

    float scale = out_delta[k + c * b] / (w * h); // Precompute scaling factor

    // Use global thread index to process each element
    for(int i = 0; i < w * h; ++i){
        int in_index = i + h * w * (k + b * c);
        in_delta[in_index] += scale; // Use precomputed scaling factor
    }
}