#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;

    // Pre-compute reused expressions
    int layer_spatial = layers * spatial;
    int in_s = index % spatial;
    int in_c = (index / spatial) % layers;
    int b = index / (layers * spatial);

    // Calculate flattened indices
    int i1 = b * layer_spatial + in_c * spatial + in_s;
    int i2 = b * layer_spatial + in_s * layers + in_c;

    // Perform read & write depending on the forward flag
    out[forward ? i2 : i1] = x[forward ? i1 : i2];
}