#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Using shared memory for biases to minimize global memory access
    __shared__ float shared_bias;
    if (threadIdx.x == 0) {
        shared_bias = biases[filter];
    }
    __syncthreads();

    if (offset < size) {
        // Eliminating unnecessary arithmetic operations inside the loop
        output[(batch * n + filter) * size + offset] *= shared_bias;
    }
}