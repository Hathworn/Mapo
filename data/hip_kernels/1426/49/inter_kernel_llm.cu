#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int blockStride = gridDim.x * blockDim.x;

    while (i < (NX + NY) * B) { // Loop over necessary indices
        int b = i / (NX + NY);
        int j = i % (NX + NY);
        if (j < NX) {
            OUT[i] = X[b * NX + j];
        } else {
            OUT[i] = Y[b * NY + j - NX];
        }
        i += blockStride; // Move to the next relevant index
    }
}