#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return; // Ensure thread index is within bounds

    float mean = 0.0f; // Local variable to accumulate mean
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            // Accumulate delta values
            mean += delta[index];
        }
    }
    // Compute and store final mean_delta result
    mean_delta[i] = mean * (-1.f / sqrtf(variance[i] + .00001f));
}