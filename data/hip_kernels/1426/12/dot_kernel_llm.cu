#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;

    // Use shared memory for better performance
    extern __shared__ float sharedData[];
    float* s_output = sharedData;
    
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            
            // Load data into shared memory for coalesced memory access
            s_output[threadIdx.x * 2] = output[i1];
            s_output[threadIdx.x * 2 + 1] = output[i2];
            __syncthreads();

            float val1 = s_output[threadIdx.x * 2];
            float val2 = s_output[threadIdx.x * 2 + 1];

            sum += val1 * val2;
            norm1 += val1 * val1;
            norm2 += val2 * val2;

            __syncthreads();
        }
    }

    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    sum = sum / norm;

    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            delta[i1] += -scale * sum * output[i2] / norm;
            delta[i2] += -scale * sum * output[i1] / norm;
        }
    }
}