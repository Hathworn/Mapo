#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + 2*pad)/stride;
    int w = (in_w + 2*pad)/stride;
    int c = in_c;
    int area = (size-1)/stride;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int index = id;
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0;
    int l, m;
    
    // Optimize memory access pattern by reducing redundant calculations within the loop
    for(l = -area; l < area+1; ++l){
        for(m = -area; m < area+1; ++m){
            int out_w = (j-w_offset + m*stride)/stride;
            int out_h = (i-h_offset + l*stride)/stride;
            int out_index = out_w + w*(out_h + h*(k + c*b));
            // Precompute validity condition to minimize branching within the loop
            bool valid = (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h);
            if(valid && indexes[out_index] == index) {
                d += delta[out_index];
            }
        }
    }
    prev_delta[index] += d;
}