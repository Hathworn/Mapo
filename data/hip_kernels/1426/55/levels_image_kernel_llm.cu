#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x;
    float s = hsv.y;
    float v = hsv.z;

    float r, g, b;
    float f, p, q, t;

    if (s == 0) {
        r = g = b = v;
    } else {
        int index = (int)h; // Optimized: floorf removed, automatic truncation
        f = h - index;
        p = v * (1 - s);
        q = v * (1 - s * f);
        t = v * (1 - s * (1 - f));

        if (index == 0) { r = v; g = t; b = p; }
        else if (index == 1) { r = q; g = v; b = p; }
        else if (index == 2) { r = p; g = v; b = t; }
        else if (index == 3) { r = p; g = q; b = v; }
        else if (index == 4) { r = t; g = p; b = v; }
        else { r = v; g = p; b = q; }
    }

    r = fminf(fmaxf(r, 0.0f), 1.0f); // Optimized: Use fminf and fmaxf for clamping
    g = fminf(fmaxf(g, 0.0f), 1.0f);
    b = fminf(fmaxf(b, 0.0f), 1.0f);

    return make_float3(r, g, b);
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x;
    float g = rgb.y;
    float b = rgb.z;

    float h, s, v;
    float max = fmaxf(fmaxf(r, g), b); // Optimized: Use fmaxf
    float min = fminf(fminf(r, g), b); // Optimized: Use fminf
    float delta = max - min;
    v = max;

    if (max != 0.0f) {
        s = delta / max;
        if (r == max) { h = (g - b) / delta; }
        else if (g == max) { h = 2.0f + (b - r) / delta; }
        else { h = 4.0f + (r - g) / delta; }
        if (h < 0.0f) h += 6.0f;
    } else {
        s = 0.0f;
        h = -1.0f;
    }
    return make_float3(h, s, v);
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Optimized: Simplified ID calculation
    int size = batch * w * h;
    if (id >= size) return;

    int x = id % w;
    int y = (id / w) % h;
    int batch_id = id / (w * h);

    float rshift = rand[0];
    float gshift = rand[1];
    float bshift = rand[2];
    float r0 = rand[8 * batch_id + 0];
    float r1 = rand[8 * batch_id + 1];
    float r2 = rand[8 * batch_id + 2];
    float r3 = rand[8 * batch_id + 3];

    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > 0.5f) ? 1.0f / saturation : saturation;
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > 0.5f) ? 1.0f / exposure : exposure;

    size_t offset = batch_id * h * w * 3;
    image += offset;
    float r = image[x + w * (y + h * 0)];
    float g = image[x + w * (y + h * 1)];
    float b = image[x + w * (y + h * 2)];
    float3 rgb = make_float3(r, g, b);

    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }

    image[x + w * (y + h * 0)] = rgb.x * scale + translate + (rshift - 0.5f) * shift;
    image[x + w * (y + h * 1)] = rgb.y * scale + translate + (gshift - 0.5f) * shift;
    image[x + w * (y + h * 2)] = rgb.z * scale + translate + (bshift - 0.5f) * shift;
}