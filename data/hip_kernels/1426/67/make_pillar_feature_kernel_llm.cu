#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void make_pillar_feature_kernel( float* dev_pillar_x_in_coors, float* dev_pillar_y_in_coors, float* dev_pillar_z_in_coors, float* dev_pillar_i_in_coors, float* dev_pillar_x, float* dev_pillar_y, float* dev_pillar_z, float* dev_pillar_i, int* dev_x_coors, int* dev_y_coors, float* dev_num_points_per_pillar, const int max_points, const int GRID_X_SIZE)
{
    // Cache number of points to reduce global memory access
    int ith_pillar = blockIdx.x;
    int num_points_at_this_pillar = dev_num_points_per_pillar[ith_pillar];
    int ith_point = threadIdx.x;

    // Exit early if the thread index exceeds the number of points
    if (ith_point >= num_points_at_this_pillar)
        return;

    // Prevent excessive recalculation of indices
    int x_ind = dev_x_coors[ith_pillar];
    int y_ind = dev_y_coors[ith_pillar];
    int pillar_base = ith_pillar * max_points;
    int coors_base = y_ind * GRID_X_SIZE * max_points + x_ind * max_points;

    int pillar_ind = pillar_base + ith_point;
    int coors_ind = coors_base + ith_point;

    // Use stored indices for memory access, reducing recalculation
    dev_pillar_x[pillar_ind] = dev_pillar_x_in_coors[coors_ind];
    dev_pillar_y[pillar_ind] = dev_pillar_y_in_coors[coors_ind];
    dev_pillar_z[pillar_ind] = dev_pillar_z_in_coors[coors_ind];
    dev_pillar_i[pillar_ind] = dev_pillar_i_in_coors[coors_ind];
}