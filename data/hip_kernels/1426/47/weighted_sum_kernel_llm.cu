#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Optimize index calculation and memory coalescing
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    if (i < n) {
        float b_val = b ? b[i] : 0.0f; // Minimize branching
        c[i] = fmaf(s[i], a[i], (1.0f - s[i]) * b_val); // Use fmaf for fused multiply-add
    }
}