#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    int stride = blockDim.x;

    local[id] = 0;
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += stride) { // Use strided loops for better memory access
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += delta[index];
        }
    }

    __syncthreads();

    // Use parallel reduction within shared memory
    for (unsigned int s = stride / 2; s > 0; s >>= 1) {
        if (id < s) {
            local[id] += local[id + s];
        }
        __syncthreads();
    }

    if (id == 0) {
        mean_delta[filter] = local[0] * (-1.f / sqrtf(variance[filter] + .00001f)); // Direct computation after reduction
    }
}