#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D grid for simplicity
    int gridSize = blockDim.x * gridDim.x; // Compute grid stride
    while (i < N) { // Loop over grid to cover all elements
        Y[i * INCY] *= X[i * INCX]; // Perform operation
        i += gridSize; // Move to the next element in the grid
    }
}