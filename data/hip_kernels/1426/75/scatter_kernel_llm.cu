#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scatter_kernel(int *x_coors, int *y_coors, float *pfe_output, float *scattered_feature, const int MAX_NUM_PILLARS_, const int GRID_X_SIZE, const int GRID_Y_SIZE)
{
    // Use blockIdx and threadIdx to calculate a unique thread index
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    int num_features = blockDim.x;  // Assuming number of features is blockDim.x

    if (threadIdx.x < num_features && blockIdx.x < MAX_NUM_PILLARS_) {
        int x_ind = x_coors[blockIdx.x];
        int y_ind = y_coors[blockIdx.x];
        float feature = pfe_output[threadIdx.x * MAX_NUM_PILLARS_ + blockIdx.x];

        int index = threadIdx.x * GRID_Y_SIZE * GRID_X_SIZE + y_ind * GRID_X_SIZE + x_ind;
        scattered_feature[index] = feature;
    }
}