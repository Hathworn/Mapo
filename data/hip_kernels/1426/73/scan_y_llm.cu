#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan_y(int *g_odata, int *g_idata, int n) 
{
    extern __shared__ int temp[];  // allocated on invocation
    int thid = threadIdx.x;
    int bid = blockIdx.x;
    int bdim = blockDim.x;
    int gdim = gridDim.x;
    int offset = 1;

    // Use a single global memory read for both elements
    int ai = bid + (2 * thid) * gdim;
    int bi = ai + gdim;
    temp[2 * thid]   = g_idata[ai]; // load input into shared memory
    temp[2 * thid+1] = g_idata[bi]; 

    // Unroll loop if possible for better performance
    for (int d = n >> 1; d > 0; d >>= 1) 
    {
        __syncthreads();
        if (thid < d) 
        {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    if (thid == 0) { temp[n - 1] = 0; } // clear the last element

    // Unroll loop if possible for down-sweep
    for (int d = 1; d < n; d *= 2) 
    {
        offset >>= 1;
        __syncthreads();
        if (thid < d) 
        {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }
    
    __syncthreads();

    // Optimize write-back by reducing conditionals
    g_odata[ai] = temp[2 * thid + 1]; // write results to device memory
    g_odata[bi] = (2 * thid + 2 == bdim * 2) ? 
                  temp[2 * thid + 1] + g_idata[bi] : 
                  temp[2 * thid + 2];
}