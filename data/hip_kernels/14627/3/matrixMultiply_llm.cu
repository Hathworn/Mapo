#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void matrixMultiply(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    // Shared memory for tiles
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    // Compute thread indices
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    float Cvalue = 0.0;

    // Loop over tiles
    for (int t = 0; t < (numAColumns-1)/TILE_WIDTH + 1; ++t) {
        if (row < numARows && t*TILE_WIDTH + threadIdx.x < numAColumns)
            tileA[threadIdx.y][threadIdx.x] = A[row*numAColumns + t*TILE_WIDTH + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0;

        if (col < numBColumns && t*TILE_WIDTH + threadIdx.y < numBRows)
            tileB[threadIdx.y][threadIdx.x] = B[(t*TILE_WIDTH + threadIdx.y)*numBColumns + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0;

        // Synchronize to ensure tiles are loaded
        __syncthreads();

        // Multiply the two tiles together
        for (int i = 0; i < TILE_WIDTH; ++i)
            Cvalue += tileA[threadIdx.y][i] * tileB[i][threadIdx.x];

        // Synchronize to ensure all threads completed multiplication
        __syncthreads();
    }

    // Write the block sub-matrix to the output matrix
    if (row < numCRows && col < numCColumns)
        C[row*numCColumns + col] = Cvalue;
}