#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Ensure thread index is within bounds
    if (idx < len) {
        // Perform vector addition
        out[idx] = in1[idx] + in2[idx];
    }
}