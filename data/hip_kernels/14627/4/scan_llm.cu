#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan(float * input, float * output, int len) {
    extern __shared__ float temp[]; // Allocate shared memory
    int tid = threadIdx.x;
    int offset = 1;
    
    // Load input into shared memory
    temp[2 * tid]     = (2 * tid < len)     ? input[2 * tid]     : 0;
    temp[2 * tid + 1] = (2 * tid + 1 < len) ? input[2 * tid + 1] : 0;

    // Up-sweep phase
    for (int d = len >> 1; d > 0; d >>= 1) {
        __syncthreads();
        if (tid < d) {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    // Clear last element
    if (tid == 0) { temp[len - 1] = 0; }

    // Down-sweep phase
    for (int d = 1; d < len; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (tid < d) {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            float t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }

    __syncthreads();
    
    // Write results to output
    if (2 * tid < len)     output[2 * tid] = temp[2 * tid];
    if (2 * tid + 1 < len) output[2 * tid + 1] = temp[2 * tid + 1];
}