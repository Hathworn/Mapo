#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
    // Obtain thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure threads do not exceed vector length
    if (idx < len) {
        // Perform vector addition
        out[idx] = in1[idx] + in2[idx];
    }
}