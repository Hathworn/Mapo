#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void matrixMultiplyShared(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    // Define shared memory for tiles
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];
    
    // Calculate the row and column index in C
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    
    float value = 0;
    // Loop over all tiles needed
    for (int i = 0; i < (numAColumns - 1) / TILE_WIDTH + 1; ++i) {
        // Load data into shared memory tiles
        if (row < numARows && (i * TILE_WIDTH + threadIdx.x) < numAColumns)
            tileA[threadIdx.y][threadIdx.x] = A[row * numAColumns + i * TILE_WIDTH + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0.0;

        if (col < numBColumns && (i * TILE_WIDTH + threadIdx.y) < numBRows)
            tileB[threadIdx.y][threadIdx.x] = B[(i * TILE_WIDTH + threadIdx.y) * numBColumns + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        // Perform multiplication for the current tile
        for (int j = 0; j < TILE_WIDTH; ++j) {
            value += tileA[threadIdx.y][j] * tileB[j][threadIdx.x];
        }
        __syncthreads();
    }

    // Write result to device memory
    if (row < numCRows && col < numCColumns) {
        C[row * numCColumns + col] = value;
    }
}