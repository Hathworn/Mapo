#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel to avoid potential race condition
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use atomic operation to safely update accuracy across threads
    atomicExch(accuracy, *accuracy / N);
}