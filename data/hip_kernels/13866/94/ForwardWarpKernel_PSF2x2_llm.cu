#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Precompute scaled flow + position
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // Break into integer and fractional parts
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    // Target pixel integer coordinates
    int tx = int(px);
    int ty = int(py);

    float value = src[image_row_offset + j];
    
    // Common condition check
    if (tx < 0 || tx >= w || ty < 0 || ty >= h) return;

    // Compute weight and update pixels
    float weight = dx * dy;
    atomicAdd(&dst[ty * image_stride + tx], value * weight);
    atomicAdd(&normalization_factor[ty * image_stride + tx], weight);

    // Bottom left: (tx-1, ty)
    if (tx > 0) {
        weight = (1.0f - dx) * dy;
        atomicAdd(&dst[ty * image_stride + (tx - 1)], value * weight);
        atomicAdd(&normalization_factor[ty * image_stride + (tx - 1)], weight);
    }

    // Upper left: (tx-1, ty-1)
    if (tx > 0 && ty > 0) {
        weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(&dst[(ty - 1) * image_stride + (tx - 1)], value * weight);
        atomicAdd(&normalization_factor[(ty - 1) * image_stride + (tx - 1)], weight);
    }

    // Upper right: (tx, ty-1)
    if (ty > 0) {
        weight = dx * (1.0f - dy);
        atomicAdd(&dst[(ty - 1) * image_stride + tx], value * weight);
        atomicAdd(&normalization_factor[(ty - 1) * image_stride + tx], weight);
    }
}