#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate linear index from 2D grid
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (i < h && j < w) {
        int pos = i * s + j; // Calculate position index

        float scale = normalization_factor[pos];
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

        image[pos] *= invScale; // Normalize the image
    }
}