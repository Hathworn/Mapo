#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit early for threads outside image bounds
    if (i < h && j < w)
    {
        // Calculate the linear index for the pixel
        const int pos = i * w + j;

        // Set the image value
        image[pos] = value;
    }
}
```
