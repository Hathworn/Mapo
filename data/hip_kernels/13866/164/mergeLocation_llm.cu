#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unrolling loop to maximize memory coalescing
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];
        x[ptidx] = __fmul_rn(loc.x, scale); // Use fast multiply
        y[ptidx] = __fmul_rn(loc.y, scale); // Use fast multiply
    }
}