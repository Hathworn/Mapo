#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Commonly reused calculation
    const int numThreads = blockDim.x * gridDim.x;

    // Use grid-stride loop for better performance
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Calculate current position in target
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Map position to source
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Assign value from source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}