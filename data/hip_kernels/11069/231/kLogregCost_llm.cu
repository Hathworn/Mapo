#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = __float2int_rd(labels[tx]);  // Use fast conversion
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);  // Fast log function

        // Optimized: use short-circuiting and early exit if not max
        if (labelp == maxp) {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp);
            }
            correctProbs[tx] = 1.0f / float(numMax);
        } else {
            correctProbs[tx] = 0.0f;
        }
    }
}