#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    // Initialize shared memory for reduction
    float sum_dY_Y = 0.0f;
    for (int i = idx; i < dim; i += blockDim.x) {
        sum_dY_Y += dY[i] * Y[i];
    }
    reduction_buffer[idx] = sum_dY_Y;
    __syncthreads();

    // Sum reduction using binary tree method
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    // Broadcast the final result
    const float reduction_result = reduction_buffer[0];

    // Compute gradient
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - reduction_result);
    }
}