#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x = ii / (d3 * d2);
    int y = (ii / d3) % d2;
    int z = (ii % d3) / scale_factor;
    int w = (off_x * scale_factor + off_y);
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x = ii / (d3 * d2);
    int y = (ii / d3) % d2;
    int z = (ii % d3) / scale_factor;
    int w = (ii % d3) / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x;
    ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
    if (ii >= no_elements) return;

    int base_idx = translate_idx(ii, d1, d2, d3, scale_factor);
    int output_idx = translate_idx_inv(base_idx, d1, d2, d3, scale_factor, 0, 0);

    // Optimize to directly fetch computed index and sum up contributions
    for (int i = 0; i < scale_factor * scale_factor; ++i) {
        gradInput_data[ii] += gradOutput_data[output_idx + i];
    }
}