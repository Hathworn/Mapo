#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Use loop unrolling for efficiency
    for (uint i = eidx; i < numElements; i += gridDim.x * DP_BLOCKSIZE) {
        sum += a[i] * b[i];
    }

    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Optimize reduction using warp-synchronous programming
    for (uint s = DP_BLOCKSIZE / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            shmem[threadIdx.x] += shmem[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}