#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    // Use thread block reduction instead of single thread for loading
    __shared__ float uni;
    if (threadIdx.x == 0) {
        uni = uniforms[blockIdx.x + blockOffset];
    }
    __syncthreads();

    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x;

    // Perform one addition safely and possibly skip the second
    if (address < n) {
        g_data[address] += uni;
    }
    if (address + blockDim.x < n) {
        g_data[address + blockDim.x] += uni;
    }
}