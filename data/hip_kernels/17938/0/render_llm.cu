#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define STB_IMAGE_WRITE_IMPLEMENTATION

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

__global__ void render(float *fb, int max_x, int max_y) {
    // Use flat 1D thread indexing for better coalesced memory access
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int pixel_index = threadId * 3;
    
    // Calculate pixel coordinates
    int i = threadId % max_x;
    int j = threadId / max_x;

    if (i < max_x && j < max_y) {
        // Directly calculate float division once and store in variable
        float inv_max_x = 1.0f / max_x;
        float inv_max_y = 1.0f / max_y;
        
        fb[pixel_index + 0] = i * inv_max_x;
        fb[pixel_index + 1] = j * inv_max_y;
        fb[pixel_index + 2] = 0.0f;
    }
}