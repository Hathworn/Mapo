#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void test(float* nonSmoothed, float* smoothed, int* mask, int nhalf) {
    // Use shared memory for low latency access
    extern __shared__ float sharedSmoothed[];

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Load smoothed value to shared memory
    if (i < nhalf) {
        sharedSmoothed[threadIdx.x] = smoothed[i];
    }
    __syncthreads();

    // Ensure index is within bounds
    if (i < nhalf) {
        // Calculate difference using shared memory value
        float diff = fabs(nonSmoothed[i] - sharedSmoothed[threadIdx.x] / nhalf);
        mask[i] = (diff > 0.23f) ? 1 : 0;   // Apply threshold
    }
}