#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void takeLog(float* input, float* env, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = i << 1;
    if (i < nhalf) {
        float value = input[j];
        // Use fmaxf to ensure logged value is always non-negative for log function
        env[i] = logf(fmaxf(value, 1e-20f));
    }
}