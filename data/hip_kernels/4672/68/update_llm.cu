#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update(float* original, float* newTE, float* current, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < nhalf) {
        float temp_current = current[i] / nhalf;  // Precompute division
        current[i] = temp_current;                // Store back result
        newTE[i] = fmaxf(original[i], temp_current); // Use intrinsic fmaxf for better performance
    }
}