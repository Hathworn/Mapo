#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expon(float* env, int nhalf) {
    // Calculate the global index based on the thread and block indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check condition to avoid out-of-bound memory access
    if (i < nhalf) {
        // Use device-specific expf function for single precision
        env[i] = expf(env[i] / nhalf);  // exponentiate
    }
}