#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update(float* original, float* newTE, float* current, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < nhalf) {
        float temp = current[i] / nhalf; // Pre-compute division to save repeated computation
        newTE[i] = fmaxf(original[i], temp); // Use fmaxf for optimized conditional selection
    }
}