#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rotatewin(float* aframe2, float *aframe, float *win, int N, int offset) {
    int k = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Check if k is within the limit and prevent out-of-bounds access
    if (k < N) {
        // Calculate the index once, avoid recalculating, for better performance
        int index = (k + offset) % N;
        aframe2[index] = win[k] * aframe[k];
    }
}