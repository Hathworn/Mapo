#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void naivekernel(float* output, float* frameA, float* frameB, int chans) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = i << 1;
    // Optimize by moving the check inside the conditional operation and using a branchless approach
    if (i < chans) {
        float a0 = frameA[j];
        float a1 = frameA[j + 1];
        float b0 = frameB[j];
        float b1 = frameB[j + 1];
        bool condition = a0 >= b0;
        output[j] = condition ? a0 : b0;
        output[j + 1] = condition ? a1 : b1;
    }
}