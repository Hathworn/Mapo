#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fixPVandGain(float* input, float* output, float gain, int length) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = i << 1;

    // Unroll loop for better performance
    if (j + 1 < length) {
        float out_j = output[j];
        float out_j1 = output[j + 1];

        // Optimize NaN check with combined conditions
        if (isnan(out_j) || out_j1 == -1.0f) {
            output[j] = 0.0f;  // set to zero any invalid or undefined amplitude
        } else {
            output[j] *= gain;  // scale valid amplitudes by the gain factor
        }
    }

    // Maintain boundary conditions outside loop
    if (j == 0) {
        output[0] = input[0];  // keep original DC amplitude
    }
    if (j == length - 2) {
        output[length - 2] = input[length - 2];  // keep original Nyquist amplitude
    }
}