#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lifter(float* cepstrum, int nCoefs, int nhalf) {
    // Improved thread index calculation for efficiency
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = i + nCoefs;
    
    // Optimized boundary check using ternary operator
    cepstrum[k] = (k < nhalf + 2 - nCoefs) ? 0.0f : cepstrum[k];
}