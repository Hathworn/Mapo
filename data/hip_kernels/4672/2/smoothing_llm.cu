#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smoothing(float* input, float* output, double alpha, double beta, int length) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Reorder for readability
    int j = i << 1; // Same as multiplying by 2
    if (j < length) {
        float in_j = input[j];      // Load input once
        float in_j1 = input[j+1];   // Load input once
        float out_j = output[j];    // Load output once
        float out_j1 = output[j+1]; // Load output once

        // Optimized calculation and store
        output[j] = (in_j * (1.0f + alpha) - out_j * alpha);
        output[j+1] = (in_j1 * (1.0f + beta) - out_j1 * beta);
    }
}