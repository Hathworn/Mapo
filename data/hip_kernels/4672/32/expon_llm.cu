#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expon(float* env, int nhalf) {
    // Optimize by using a grid stride loop to ensure all elements are processed
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < nhalf; i += blockDim.x * gridDim.x) {
        env[i] = exp(env[i] / nhalf); // Exponentiate
    }
}