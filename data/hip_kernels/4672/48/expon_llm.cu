#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expon(float* env, int nhalf) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop unrolling to increase efficiency
    if (i < nhalf) {
        env[i] = expf(env[i] / nhalf); // Use expf for single precision
        if (i + blockDim.x < nhalf) {
            env[i + blockDim.x] = expf(env[i + blockDim.x] / nhalf);
        }
    }
}