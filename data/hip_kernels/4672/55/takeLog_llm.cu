#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void takeLog(float* input, float* env, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < nhalf) {
        int j = i << 1;
        // Use fmaxf to ensure input[j] is not less than 1e-20 before taking log
        env[i] = logf(fmaxf(input[j], 1e-20f)); 
    }
}