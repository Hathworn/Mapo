#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update(float* original, float* newTE, float* current, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Ensure 'i' is within bounds before continuing computations
    if (i < nhalf) {
        current[i] /= nhalf;

        // Avoid branch by using fmaxf, which computes the maximum of two values
        newTE[i] = fmaxf(original[i], current[i]); 
    }
}