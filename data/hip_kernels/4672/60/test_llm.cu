#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void test(float* nonSmoothed, float* smoothed, int* mask, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < nhalf) {
        // Precompute the smoothed value division
        float smoothVal = smoothed[i] / nhalf;
        // Use a ternary operator with fabs to compute the mask value
        mask[i] = (fabs(nonSmoothed[i] - smoothVal) > 0.23f) ? 1 : 0;
    }
}