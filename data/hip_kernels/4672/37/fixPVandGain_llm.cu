#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fixPVandGain(float* output, float gain, int length) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = i << 1;
    if (j < length) {
        // Replace isnan with inline check to reduce function call overhead
        if (!(output[j] == output[j])) // Check for NaN
            output[j] = 0.0f; // Set to zero any invalid amplitude

        float freq = output[j + 1];
        if (freq == -1.0f) { // Check for undefined frequency
            output[j] = 0.0f; // Set to zero the amp related to any undefined frequency
        } else {
            output[j] *= gain; // Scale all amplitudes by the gain factor
        }
    }
}