#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expon(float* env, int nhalf) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use improved branch divergence
    if (i >= nhalf) return;

    // Precompute the division for better performance
    float invNhalf = 1.0f / nhalf;
    env[i] = exp(env[i] * invNhalf);
}