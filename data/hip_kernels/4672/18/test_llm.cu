#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void test(float* nonSmoothed, float* smoothed, int* mask, int nhalf) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nhalf) {
        // Use local variables to reduce global memory access
        float nonSmoothVal = nonSmoothed[i];
        float smoothVal = smoothed[i];
        
        // Calculate difference with potential optimization
        float diff = fabs(nonSmoothVal - smoothVal / nhalf);
        
        // Assign mask value based on threshold
        mask[i] = (diff > 0.23f) ? 1 : 0;  // Ensure to include 'f' to specify float literal
    }
}