#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void morph(float* output, float* input1, float* input2, float ampCoeff, float freqCoeff, int length) {
    int i = (blockIdx.x * blockDim.x + threadIdx.x) << 1; // Combine index calculation
    if (i < length) {
        float amp = 1.0 - ampCoeff;  // Pre-calculate coefficients
        float freq = 1.0 - freqCoeff;
        
        // Load input values into registers for faster access
        float in1_0 = input1[i]; 
        float in1_1 = input1[i + 1];
        float in2_0 = input2[i];
        float in2_1 = input2[i + 1];
        
        // Compute output using pre-calculated coefficients
        output[i] = in1_0 * amp + in2_0 * ampCoeff;
        output[i + 1] = in1_1 * freq + in2_1 * freqCoeff;
    }
}