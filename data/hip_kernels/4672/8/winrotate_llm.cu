#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Global kernel function
__global__ void winrotate(float* inframe2, const float* inframe, const float *win, int N, int offset) {
    int k = threadIdx.x + blockIdx.x * blockDim.x;
    if (k < N) {
        // Perform computation only if within bounds
        int index = (k + offset) % N;
        inframe2[k] = win[k] * inframe[index];
    }
}