#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update(float* original, float* newTE, float* current, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < nhalf) {
        // Pre-compute the reciprocal to avoid division in the loop
        float inv_nhalf = 1.0f / nhalf;
        
        // Use reciprocal multiplication instead of division
        float curr_val = current[i] * inv_nhalf;
        
        // Simplified the conditional expression by using max function
        newTE[i] = fmaxf(original[i], curr_val);
    }
}