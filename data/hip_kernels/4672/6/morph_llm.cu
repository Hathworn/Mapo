#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void morph(float* output, float* input1, float* input2, float ampCoeff, float freqCoeff, int length) {
    int j = (threadIdx.x + blockIdx.x * blockDim.x) << 1;
    // Optimize the condition check by moving it to an earlier stage
    if (j < length) {
        float ampCoeffComp = 1.0f - ampCoeff;
        float freqCoeffComp = 1.0f - freqCoeff;
        // Reduce redundant computations
        output[j] = input1[j] * ampCoeffComp + input2[j] * ampCoeff;
        output[j + 1] = input1[j + 1] * freqCoeffComp + input2[j + 1] * freqCoeff;
    }
}