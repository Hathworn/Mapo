#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update(float* original, float* newTE, float* current, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < nhalf) {
        // Use reciprocal for division to improve performance
        float reciprocal_nhalf = 1.0f / nhalf;
        current[i] *= reciprocal_nhalf; 
        // Use ternary operator efficiently to avoid branching
        newTE[i] = max(original[i], current[i]);
    }
}