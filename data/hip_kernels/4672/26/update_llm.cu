#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update(float* original, float* newTE, float* current, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < nhalf) {
        current[i] /= nhalf;
        // Use max function to improve performance
        newTE[i] = max(original[i], current[i]);
    }
}