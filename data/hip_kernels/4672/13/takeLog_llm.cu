#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void takeLog(float* input, float* env, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < nhalf) {
        int j = i << 1; // Separate index calculation to avoid redundant operations
        float val = input[j]; // Load input value once
        env[i] = log(fmaxf(val, 1e-20f)); // Use fmaxf for streamlined comparison and single function call
    }
}