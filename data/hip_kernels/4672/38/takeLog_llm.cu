#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void takeLog(float* input, float* env, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = i << 1;
    
    // Efficient check within thread limit for better performance
    if (i < nhalf) {
        // Avoid branching by using the ternary operator directly in computation
        float value = input[j];
        env[i] = logf(fmaxf(value, 1e-20f));  // Take the log safely with built-in functions
    }
}