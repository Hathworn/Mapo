#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fixPVandGain(float* output, float gain, int lowestIndx, int length) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = i << 1;

    if (j >= lowestIndx && j < length) {
        float amplitude = output[j];
        if (__builtin_isnan(amplitude)) {  // Use intrinsic function for better performance
            amplitude = 0.0f;  // set to zero any invalid amplitude
        } else if (output[j + 1] == -1.0f) {
            amplitude = 0.0f;  // set to zero the amp related to any undefined frequency
        } else {
            amplitude *= gain;  // scale all amplitudes by the gain factor
        }
        output[j] = amplitude;  // Ensure output update outside of conditional logic
    }
}