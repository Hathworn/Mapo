#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void takeLog(float* input, float* env, int nhalf) {
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int dataIndex = globalIndex << 1;
    
    // Process only within the valid range
    if (globalIndex < nhalf) {
        float value = input[dataIndex];
        // Use fmaxf for better performance with small values
        env[globalIndex] = logf(fmaxf(value, 1e-20f));
    }
}