#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lifter(float* cepstrum, int nCoefs, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // Calculate k once and reuse.
    int k = i + nCoefs;
    
    // Bounds checking to avoid out-of-bound access
    if (k < nhalf + 2 - nCoefs) {
        cepstrum[k] = 0.0; // Efficiently kill all the cepstrum coefficients above nCoefs
    }
}