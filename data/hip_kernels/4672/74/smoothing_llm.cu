#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smoothing(float* input, float* output, double alpha, double beta, int length) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = i << 1;
    if (j < length) {
        // Reduce memory accesses by reusing computed values
        float input_j = input[j];
        float input_j1 = input[j+1];
        float output_j = output[j];
        float output_j1 = output[j+1];
        
        // Use constant expressions to minimize repeated calculations
        double alpha_term = 1.0 + alpha;
        double beta_term = 1.0 + beta;

        output[j] = (float) (input_j * alpha_term - output_j * alpha);
        output[j+1] = (float) (input_j1 * beta_term - output_j1 * beta);
    }
}