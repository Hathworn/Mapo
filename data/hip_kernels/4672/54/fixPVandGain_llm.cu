#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fixPVandGain(float* input, float* output, float gain, int lowestIndx, int length) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = i << 1;
    
    // Use early return to reduce unnecessary checks and improve performance
    if (j < lowestIndx || j >= length) return;

    // Replace isnan with fpclassify for potential performance gain
    if (__builtin_fpclassify(FP_NAN, FP_INFINITE, FP_NORMAL, FP_SUBNORMAL, FP_ZERO, output[j]) == FP_NAN) 
        output[j] = 0.0f;  // Set to zero any invalid amplitude

    if (output[j+1] == -1.0f) 
        output[j] = 0.0f;  // Set to zero the amp related to any undefined frequency
    else
        output[j] *= gain;  // Scale all amplitudes by the gain factor

    if (j == 0) output[0] = input[0];  // Keep original DC amplitude
    if (j == length-2) output[length-2] = input[length-2];  // Keep original Nyquist amplitude
}