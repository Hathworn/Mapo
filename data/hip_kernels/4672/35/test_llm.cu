#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void test(float* nonSmoothed, float* smoothed, int* mask, int nhalf) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    if (i < nhalf) {
        float diff = fabsf(nonSmoothed[i] - smoothed[i] / nhalf);  // Use fabsf for faster single-precision
        mask[i] = (diff > 0.23f) ? 1 : 0;  // Use 0.23f for consistency
    }
}