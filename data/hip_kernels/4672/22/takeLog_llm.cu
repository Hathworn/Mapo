#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void takeLog(float* input, float* env, int nhalf) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure i is within valid range
    if (i < nhalf) {
        int j = i << 1; // Use bit shift for multiplication by 2
        float amplitude = input[j];
        // Use fmax to ensure amplitude is greater than 0
        env[i] = logf(fmaxf(amplitude, 1e-20f)); // Use logf for faster calculations
    }
}