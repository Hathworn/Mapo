#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lifter(float* cepstrum, int nCoefs, int nhalf) {
    int k = threadIdx.x + blockDim.x * blockIdx.x + nCoefs;
    if (k < nhalf + 2) {  // Simplify condition without recalculating
        cepstrum[k] = 0.0f; // Use '0.0f' for float
    }
}