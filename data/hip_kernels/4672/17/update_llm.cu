#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update(float* original, float* newTE, float* current, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < nhalf) {
        float cur = current[i];
        cur /= nhalf;  // Reduce repeated division, improves performance
        current[i] = cur;
        float orig = original[i];
        newTE[i] = (orig < cur) ? cur : orig;  // Eliminates multiple memory accesses
    }
}