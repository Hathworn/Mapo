#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lifter(float* cepstrum, int nCoefs, int nhalf) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = i + nCoefs;
    if (k < nhalf + 2 - nCoefs) {
        // Set to zero for coefficients above nCoefs
        cepstrum[k] = 0.0f;
    }
}