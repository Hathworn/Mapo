#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fixPVandGain(float* input, float* output, float gain, int length) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = i << 1;

    // Use __ldg for potentially optimized global memory access
    if (j < length) {
        // Combine condition checks to reduce branching
        float out_j = __ldg(&output[j]);
        if (isnan(out_j) || output[j + 1] == -1.0f) {
            output[j] = 0.0f; // set to zero any invalid or undefined frequency amplitude
        } else {
            output[j] *= gain; // scale all amplitudes by the gain factor
        }
    }

    // Use ternary operator to simplify boundary conditions
    if (j == 0 || j == length - 2) {
        output[j] = (j == 0) ? input[0] : input[length - 2]; // keep original DC or Nyquist amplitude
    }
}