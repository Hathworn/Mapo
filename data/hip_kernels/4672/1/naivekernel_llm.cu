#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void naivekernel(float* output, float* frameA, float* frameB, int chans) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = i << 1;
    
    // Check if index is within bounds before accessing memory
    if (i < chans) {
        float2 a = reinterpret_cast<float2*>(frameA)[i]; // Load two floats at once
        float2 b = reinterpret_cast<float2*>(frameB)[i]; // Load two floats at once
        
        // Use inline conditional operator for concise code
        reinterpret_cast<float2*>(output)[i] = (a.x >= b.x) ? a : b;
    }
}