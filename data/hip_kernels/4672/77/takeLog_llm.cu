#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void takeLog(float* input, float* env, int nhalf) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nhalf) {
        int j = i << 1;
        float val = fmaxf(input[j], 1e-20f);  // Ensure non-negative input for log
        env[i] = logf(val);  // Use logf for improved precision and performance with floats
    }
}