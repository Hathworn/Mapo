#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lifter(float* cepstrum, int nCoefs, int nhalf) {
    int k = threadIdx.x + blockDim.x * blockIdx.x + nCoefs;
    if (k < nhalf + 2 - nCoefs) {
        cepstrum[k] = 0.0; // Set unused cepstrum coefficients to zero
    }
}