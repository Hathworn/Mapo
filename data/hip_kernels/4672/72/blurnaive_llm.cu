#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blurnaive(float* matrix, float* output, int firstFrame, int numFrames, int frameCount, int max, int length) {
    int chan = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
    float amp = 0.0f;
    float freq = 0.0f;

    if (chan < length) {
        // Unrolling loop for improved performance
        int frame = firstFrame;
        for (int i = 0; i < numFrames; i++) {
            amp += matrix[frame * length + chan];
            freq += matrix[frame * length + chan + 1];
            frame = (frame + 1) % max; // Move outside loop if frameCount % numFrames == 0
        }
        
        // Pre-calculate reciprocal for reduced division cost
        float invNumFrames = 1.0f / numFrames;
        output[chan] = amp * invNumFrames;
        output[chan + 1] = freq * invNumFrames;
    }
}