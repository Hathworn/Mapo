#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expon(float* env, int nhalf) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Prevent threads from processing out of range data
    if (i >= nhalf) return;

    // Improved exponentiation operation with better memory access pattern
    const float inv_nhalf = 1.0f / nhalf;
    env[i] = exp(env[i] * inv_nhalf);
}