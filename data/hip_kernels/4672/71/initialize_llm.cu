#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void initialize(float* matrix, float sr, int numFrames, int length) {
    int frame = blockIdx.y * blockDim.y + threadIdx.y;
    int chan  = (blockIdx.x * blockDim.x + threadIdx.x) << 1;
    if (frame < numFrames && chan < length) {
        int index = frame * length + chan; // Calculate index once
        matrix[index] = 0.0f;
        matrix[index + 1] = chan * sr / (length - 2.0f); // Use float division for consistency
    }
}