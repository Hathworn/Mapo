#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blurnaive(float* matrix, float* output, int firstFrame, int numFrames, int frameCount, int max, int length) {
    int chan = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
    float amp = 0.0f;
    float freq = 0.0f;
    if (chan < length) {
        // Use shared memory for reduced global memory access
        extern __shared__ float shared_data[];
        float* shared_matrix = shared_data;
        
        for (int frame = firstFrame; frame != frameCount; frame = (frame + 1) % max) {
            // Load matrix elements into shared memory
            shared_matrix[threadIdx.x * 2] = matrix[frame * length + chan];
            shared_matrix[threadIdx.x * 2 + 1] = matrix[frame * length + chan + 1];
            __syncthreads(); // Ensure all threads have loaded the data
            
            // Accumulate summations using the shared memory
            amp += shared_matrix[threadIdx.x * 2];
            freq += shared_matrix[threadIdx.x * 2 + 1];
            __syncthreads();
        }
        output[chan] = amp / numFrames;
        output[chan + 1] = freq / numFrames;
    }
}