#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lifter(float* cepstrum, int nCoefs, int nhalf) {
    int k = blockIdx.x * blockDim.x + threadIdx.x + nCoefs;

    // Check and compute only when k is in range
    if (k < nhalf + 2) {
        cepstrum[k] = 0.0;
    }
}