#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expon(float* env, int nhalf) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use loop unrolling for improved performance
    for (; i < nhalf; i += stride) {
        env[i] = expf(env[i] / nhalf); // Use expf for single precision
    }
}