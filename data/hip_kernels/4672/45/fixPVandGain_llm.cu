#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fixPVandGain(float* output, float gain, int lowestIndx, int length) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = i << 1;
    if (j >= lowestIndx && j < length) {
        float val = output[j];
        
        // Optimize isnan check by using a bitwise trick
        if (!(val == val)) { 
            val = 0.0f;   // set to zero any invalid amplitude
        } else {
	        // Combine checks for output[j+1]
            float nextVal = output[j + 1];
            if (nextVal != -1.0f) {
                val *= gain; // scale all amplitudes by the gain factor
            } else {
                val = 0.0f;  // set to zero the amp related to any undefined frequency
            }
        }
        
        output[j] = val;
    }
}