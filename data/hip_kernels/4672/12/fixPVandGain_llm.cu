#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fixPVandGain(float* output, float gain, int length) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = i << 1;
    if (j < length) {
        float val = output[j];
        float freq = output[j+1];
        if (isnan(val) || freq == -1.0f) { // Combine checks to reduce branching
            output[j] = 0.0f;  // Set to zero for invalid conditions
        } else {
            output[j] *= gain; // Scale amplitudes by gain factor
        }
    }
}