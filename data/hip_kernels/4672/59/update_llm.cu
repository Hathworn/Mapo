#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update(float* original, float* newTE, float* current, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < nhalf) {
        current[i] /= nhalf;
        // Use ternary operator efficiently by assigning directly to newTE[i]
        newTE[i] = fmaxf(original[i], current[i]);
    }
}