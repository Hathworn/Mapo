#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lifter(float* cepstrum, int nCoefs, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int k = i + nCoefs;
    
    // Use block size to ensure we process within the valid range
    if (k < nhalf + 2 - nCoefs) {
        cepstrum[k] = 0.0f;  // Set coefficients beyond nCoefs to 0
    }
}