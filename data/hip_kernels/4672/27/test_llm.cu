#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void test(float* nonSmoothed, float* smoothed, int* mask, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < nhalf) {
        // Store smoothed[i] divided by nhalf in a temporary variable to avoid redundant division
        float smoothVal = smoothed[i] / nhalf;
        
        // Use fabsf for single precision floating point absolute value
        float diff = fabsf(nonSmoothed[i] - smoothVal);

        // Apply thresholding
        mask[i] = (diff > 0.23f) ? 1 : 0;
    }
}