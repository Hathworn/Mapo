#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fixPVandGain(float* output, float gain, int length) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = i << 1;
    
    // Check boundaries first to avoid unnecessary calculations
    if (j < length) {
        float val_j = output[j];
        float val_j1 = output[j+1];

        // Combine logical checks to minimize divergence
        if (!isnan(val_j) && val_j1 != -1.0f) {
            output[j] *= gain; // scale if both checks pass
        } else {
            output[j] = 0.0f; // set to zero for invalid or undefined conditions
        }
    }
}