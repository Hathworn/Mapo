#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void test(float* nonSmoothed, float* smoothed, int* mask, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x; // Calculate global thread index
    if (i < nhalf) {
        float smoothedVal = smoothed[i] / nhalf; // Precompute smooth value
        int diff = fabsf(nonSmoothed[i] - smoothedVal); // Use fabsf for float
        mask[i] = (diff > 0.23f) ? 1 : 0; // Use float literal for threshold
    }
}