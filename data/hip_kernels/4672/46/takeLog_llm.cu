#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void takeLog(float* input, float* env, int nhalf) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nhalf) {
        float value = input[i << 1];               // Optimized: Pre-compute the index for clarity
        env[i] = logf(fmaxf(value, 1e-20f));       // Optimized: Use logf and fmaxf for float precision
    }
}