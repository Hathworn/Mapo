#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expon(float* env, int nhalf) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Loop unrolling
    int stride = blockDim.x * gridDim.x;
    for (int idx = i; idx < nhalf; idx += stride) {
        env[idx] = __expf(env[idx] / nhalf); // Use fast math
    }
}