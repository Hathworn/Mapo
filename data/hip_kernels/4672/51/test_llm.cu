#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void test(float* nonSmoothed, float* smoothed, int* mask, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Launch fewer threads and check within bounds
    if (i >= nhalf) return;

    float diff = fabs(nonSmoothed[i] - smoothed[i] / nhalf);
    mask[i] = (diff > 0.23f) ? 1 : 0; // Use f for float literal
}