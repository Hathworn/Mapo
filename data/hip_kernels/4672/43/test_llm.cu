#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void test(float* nonSmoothed, float* smoothed, int* mask, int nhalf) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // Ensure the computation is within bounds
    if (i < nhalf) {
        // Cache computation to avoid recalculating
        float smoothedValue = smoothed[i] / nhalf;
        float diff = fabs(nonSmoothed[i] - smoothedValue);
        
        // Use ternary operator for concise assignment
        mask[i] = (diff > 0.23f) ? 1 : 0;
    }
}