#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expon(float* env, int nhalf) {
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Process only necessary elements within bounds
    if (i < nhalf) {
        // Exponentiate efficiently
        env[i] = expf(env[i] / nhalf);
    }
}