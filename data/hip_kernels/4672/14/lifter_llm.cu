#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lifter(float* cepstrum, int nCoefs, int nhalf) {
    int k = threadIdx.x + blockDim.x * blockIdx.x + nCoefs;
    if (k < nhalf + 2) { // Simplify condition
        cepstrum[k] = 0.0f; // Use float literal
    }
}