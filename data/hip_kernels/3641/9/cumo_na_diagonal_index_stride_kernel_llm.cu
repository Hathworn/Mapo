#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cumo_na_diagonal_index_stride_kernel(size_t *idx, size_t *idx0, ssize_t s1, size_t k0, size_t k1, uint64_t n)
{
    // Use a loop stride technique to improve memory access patterns
    uint64_t stride = blockDim.x * gridDim.x;
    uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    while (i < n) {
        idx[i] = idx0[i + k0] + s1 * (i + k1);
        i += stride; // Increment by the total number of threads
    }
}