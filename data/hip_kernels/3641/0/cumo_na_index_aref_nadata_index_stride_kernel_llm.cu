#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cumo_na_index_aref_nadata_index_stride_kernel(size_t *idx, ssize_t s1, uint64_t n)
{
    // Calculate global index once and reuse variable
    uint64_t global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop without recalculating thread index per iteration
    for (uint64_t i = global_idx; i < n; i += blockDim.x * gridDim.x) {
        idx[i] *= s1;  // Direct multiplication assignment
    }
}