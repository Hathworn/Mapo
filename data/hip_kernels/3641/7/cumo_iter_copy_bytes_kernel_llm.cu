#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cumo_iter_copy_bytes_kernel(char *p1, char *p2, ssize_t s1, ssize_t s2, size_t *idx1, size_t *idx2, uint64_t n, ssize_t elmsz)
{
    // Calculate the global thread index
    uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop through the index space in strides of total number of threads
    while (i < n) {
        // Compute source and destination addresses
        char *p1_ = p1 + (idx1 ? idx1[i] : i * s1);
        char *p2_ = p2 + (idx2 ? idx2[i] : i * s2);
        // Perform memory copy
        memcpy(p2_, p1_, elmsz);
        // Update the global index for the next iteration
        i += blockDim.x * gridDim.x;
    }
}