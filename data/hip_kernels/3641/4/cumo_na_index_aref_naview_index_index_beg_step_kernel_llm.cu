#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cumo_na_index_aref_naview_index_index_beg_step_kernel(size_t *idx, size_t *idx1, size_t beg, ssize_t step, uint64_t n)
{
    // Compute this thread's unique global index
    uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop pattern for better memory access pattern and load balancing
    uint64_t stride = blockDim.x * gridDim.x;
    while (i < n) {
        idx[i] = idx1[beg + step * i];
        i += stride;
    }
}