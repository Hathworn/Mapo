#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cumo_na_index_aref_naview_index_stride_kernel(size_t *idx, ssize_t s1, uint64_t n)
{
    // Calculate the global thread index
    uint64_t i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop with stride to process all elements
    for (; i < n; i += blockDim.x * gridDim.x) {
        idx[i] *= s1; // Perform the multiplication
    }
}