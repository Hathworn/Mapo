#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cumo_na_index_aref_naview_index_stride_last_kernel(size_t *idx, ssize_t s1, size_t last, uint64_t n)
{
    // Calculate global thread ID
    uint64_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop pattern for better performance
    for (uint64_t i = tid; i < n; i += blockDim.x * gridDim.x) {
        idx[i] = (last - idx[i]) * s1;
    }
}