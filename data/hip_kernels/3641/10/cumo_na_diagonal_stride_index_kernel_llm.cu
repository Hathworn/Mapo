#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cumo_na_diagonal_stride_index_kernel(size_t *idx, ssize_t s0, size_t *idx1, size_t k0, size_t k1, uint64_t n)
{
    // Calculate unique thread index within the grid
    uint64_t threadId = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure only active threads perform calculations
    if (threadId < n) {
        idx[threadId] = s0 * (threadId + k0) + idx1[threadId + k1];
    }
}