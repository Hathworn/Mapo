#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Using thread index to ensure operation is done by only one thread
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}