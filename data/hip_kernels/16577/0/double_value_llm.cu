#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void double_value(double *x, double *y)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Double the value at the global index
    y[idx] = 2.0 * x[idx];
}