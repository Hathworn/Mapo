#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    extern __shared__ float shared_data[];  // Shared memory for reduction
    float *s_input = shared_data;
    float *s_output = &shared_data[n];
    
    int tid = threadIdx.x;
    float sum = 0;
    float largest = -INFINITY;
    
    // Use multiple threads to find the largest value
    for (int i = tid; i < n; i += blockDim.x) {
        float val = input[i*stride];
        s_input[i] = val;
        atomicMax((int*)&largest, __float_as_int(val));
    }
    __syncthreads();
    
    // Compute exponentials and sum
    for (int i = tid; i < n; i += blockDim.x) {
        float e = expf(s_input[i] / temp - largest / temp);
        sum += e;
        s_output[i] = e;
    }
    __syncthreads();
    
    // Use atomicAdd to accumulate the sum across threads
    atomicAdd(&s_output[0], sum);
    __syncthreads();
    
    sum = s_output[0];
    
    // Normalize output
    for (int i = tid; i < n; i += blockDim.x) {
        output[i * stride] = s_output[i] / sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= spatial*batch*groups) return;
    
    int s = id % spatial;
    id = id / spatial;
    int g = id % groups;
    int b = id / groups;
    int goff = group_offset[g] * spatial;
    int boff = b * stride;

    // Launch softmax device function with shared memory
    int n = group_size[g];
    softmax_device<<<1, min(n, 1024), 2 * n * sizeof(float)>>>(input + goff + boff + s, n, temp, spatial, output + goff + boff + s);
}