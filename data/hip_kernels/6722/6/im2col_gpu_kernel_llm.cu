#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void im2col_gpu_kernel(const int n, const float* data_im, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_col) {
    // Precompute some values outside the loop for efficiency
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int width_range = width_col * height_col;
    
    for (; idx < n; idx += blockDim.x * gridDim.x) {
        int w_out = idx % width_col;
        int h_index = idx / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;

        float* data_col_ptr = data_col + (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im + (channel_in * height + h_in) * width + w_in;
        
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                // Conditional efficiently checks bounds and assigns values
                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ? data_im_ptr[i * width + j] : 0;

                data_col_ptr += width_range; // Use precomputed width_range for pointer increment
            }
        }
    }
}