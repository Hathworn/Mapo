#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    extern __shared__ float shared_mem[];
    float *e_array = shared_mem;
    
    int tid = threadIdx.x;
    float sum = 0.0f;
    float largest = -INFINITY;

    // Determine the number of elements per thread
    int elements_per_thread = (n + blockDim.x - 1) / blockDim.x;

    // Calculate the largest value with parallel reduction
    for (int i = 0; i < elements_per_thread; ++i) {
        int index = tid + i * blockDim.x;
        if (index < n) {
            float val = input[index * stride];
            atomicMax(&largest, val);
        }
    }
    __syncthreads();

    // Calculate the exponentials and sum with parallel reduction
    for (int i = 0; i < elements_per_thread; ++i) {
        int index = tid + i * blockDim.x;
        if (index < n) {
            float e = expf(input[index * stride] / temp - largest / temp);
            e_array[index] = e;
            atomicAdd(&sum, e);
        }
    }
    __syncthreads();

    // Normalize output
    for (int i = 0; i < elements_per_thread; ++i) {
        int index = tid + i * blockDim.x;
        if (index < n) {
            output[index * stride] = e_array[index] / sum;
        }
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= batch * groups) return;

    int b = id / groups;
    int g = id % groups;
    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}