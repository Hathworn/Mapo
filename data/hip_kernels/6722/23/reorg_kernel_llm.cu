#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate the unique thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= N) return;

    int in_index = index;
    int in_w = index % w;
    int in_h = (index / w) % h;
    int in_c = (index / (w * h)) % c;
    int b = index / (w * h * c);
    
    int out_c = c / (stride * stride);
    int offset = in_c / out_c;
    int c2 = in_c % out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Perform the read/write operation
    if(forward) 
        out[out_index] = x[in_index];
    else 
        out[in_index] = x[out_index];
}