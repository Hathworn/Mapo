#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[BLOCK];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0;
    float sum = 0;

    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            sum += delta[index] * (x[index] - mean[filter]);  // Precompute index once.
        }
    }

    local[id] = sum;  // Accumulate partial sum results.

    __syncthreads();

    // Reduce sums within a block.
    for (int s = threads / 2; s > 0; s >>= 1) {
        if (id < s) {
            local[id] += local[id + s];
        }
        __syncthreads();
    }

    if (id == 0) {
        variance_delta[filter] = local[0] * -.5f * powf(variance[filter] + .00001f, -1.5f);
    }
}