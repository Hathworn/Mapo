#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use shared memory for faster data access
    extern __shared__ float shared_X[];

    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    
    // Load data into shared memory
    if (i < N) {
        shared_X[threadIdx.x] = X[i * INCX];
        __syncthreads();
        
        // Perform computation using shared memory
        if ((shared_X[threadIdx.x] * shared_X[threadIdx.x]) < (ALPHA * ALPHA)) {
            X[i * INCX] = 0;
        }
    }
}