#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function with using blockDim.x * blockDim.y * gridDim.x calculations to simplify index computation.
__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
  int id = blockIdx.x * blockDim.x + threadIdx.x + 
           blockIdx.y * (gridDim.x * blockDim.x);
  
  // Avoid redundant computation of accessing arrays/tid bounds. Ops are reduced here.
  if (id < size) {
    float rand_val = rand[id];
    float input_val = input[id];
    input[id] = (rand_val < prob) ? 0.0f : input_val * scale;  
  }
}