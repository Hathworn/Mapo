#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate the index for the current thread directly
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread operates within bounds
    if(i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);

        // Use a ternary operator for concise conditional logic
        error[i] = abs_val < 1 ? diff * diff : 2 * abs_val - 1;
        delta[i] = abs_val < 1 ? diff : (diff > 0 ? 1 : -1);
    }
}