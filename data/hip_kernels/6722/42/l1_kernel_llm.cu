#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if(index < n){
        float diff = truth[index] - pred[index]; // Use single index calculation
        error[index] = fabsf(diff); // Use CUDA intrinsic function for absolute value
        delta[index] = copysignf(1.0f, diff); // Use copysignf for sign evaluation
    }
}