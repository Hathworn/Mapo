#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use strided access pattern
    for (int idx = i; idx < N; idx += gridDim.x * blockDim.x) {
        X[idx * INCX] *= ALPHA;
    }
}