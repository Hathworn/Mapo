#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes) {
    int h = (in_h + 2*pad)/stride;
    int w = (in_w + 2*pad)/stride;
    int c = in_c;
    int area = (size-1)/stride;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int index = id;
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0;

    // Use a single loop to optimize the iteration over the pooling window
    for(int l = -area; l <= area; ++l){
        int out_h = (i - h_offset) / stride + l;
        if(out_h < 0 || out_h >= h) continue; // Check boundaries for better performance

        for(int m = -area; m <= area; ++m){
            int out_w = (j - w_offset) / stride + m;
            if(out_w < 0 || out_w >= w) continue; // Check boundaries for better performance

            int out_index = out_w + w*(out_h + h*(k + c*b));
            if (indexes[out_index] == index) {
                d += delta[out_index]; // Apply delta only when index matches
            }
        }
    }

    prev_delta[index] += d;
}