#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified calculation of index within 1D grid
    if (i < n && mask[i] == mask_num) // Check if within bounds
    {
        x[i] = mask_num; // Assign value based on mask
    }
}