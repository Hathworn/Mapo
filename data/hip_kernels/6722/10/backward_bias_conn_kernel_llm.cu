#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    // Use shared memory to store partial sums for a block
    extern __shared__ float shared_sum[];
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    
    if (index < n) {
        float sum = 0;
        for (int b = 0; b < batch; ++b) {
            int i = b * n + index;
            sum += delta[i];
        }
        shared_sum[tid] = sum;
    } else {
        shared_sum[tid] = 0;
    }

    __syncthreads();

    // Perform parallel reduction within the block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_sum[tid] += shared_sum[tid + stride];
        }
        __syncthreads();
    }

    // Write the result to global memory (one thread per block writes)
    if (tid == 0) {
        atomicAdd(&bias_updates[blockIdx.x], shared_sum[0]);
    }
}