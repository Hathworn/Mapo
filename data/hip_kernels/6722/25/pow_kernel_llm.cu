#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate grid stride

    // Loop with grid-stride to process larger datasets
    for (; i < N; i += stride)
    {
        Y[i * INCY] = powf(X[i * INCX], ALPHA); // Use powf for float precision
    }
}