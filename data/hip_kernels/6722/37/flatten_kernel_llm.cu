#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate the global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within bounds
    if (idx < N) {
        int b = idx / (layers * spatial);     // Batch index
        int i = idx % (layers * spatial);     // Remaining index after batch calculation
        int in_c = i / spatial;               // Layer index
        int in_s = i % spatial;               // Spatial index

        // Compute linear indices for input and output based on the forward flag
        int in_index = b * layers * spatial + in_c * spatial + in_s;
        int out_index = forward ? (b * layers * spatial + in_s * layers + in_c) 
                                : in_index;
        
        if (forward) {
            out[out_index] = x[in_index];    // Copy data in forward direction
        } else {
            out[in_index] = x[out_index];    // Copy data in backward direction
        }
    }
}