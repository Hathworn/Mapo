#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (index < N) { // Inverted condition to reduce branching
        int f = (index / spatial) % filters;
        x[index] = (x[index] - mean[f]) / __fsqrt_rn(variance[f] + .00001f); // Use intrinsic function for faster sqrt
    }
}