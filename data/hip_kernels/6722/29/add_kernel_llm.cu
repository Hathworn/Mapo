#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the unique thread index in the grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Process corresponding elements only within bounds
    if (i < N) {
        X[i * INCX] += ALPHA;
    }
}