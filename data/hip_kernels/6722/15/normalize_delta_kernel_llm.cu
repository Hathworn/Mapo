#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is within bounds
    if (index < N) {
        int f = (index / spatial) % filters;
        float inv_sqrt_var = rsqrtf(variance[f] + 1e-5f); // Use fast rsqrt function
        float var_delta_term = variance_delta[f] * 2.0f * (x[index] - mean[f]) / (spatial * batch);
        float mean_delta_term = mean_delta[f] / (spatial * batch);
        
        // Update delta with optimized computation
        delta[index] = delta[index] * inv_sqrt_var + var_delta_term + mean_delta_term;
    }
}