#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Declare shared memory for mean_delta
    extern __shared__ float shared_mean_delta[];

    shared_mean_delta[threadIdx.x] = 0;
    
    // Accumulate in shared memory
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
             int index = j * filters * spatial + i * spatial + k;
             shared_mean_delta[threadIdx.x] += delta[index];
        }
    }

    // Synchronize threads to ensure completion of shared memory operations
    __syncthreads();

    // Transfer results to global memory
    if (threadIdx.x == 0) {
        float block_sum = 0;
        for (int j = 0; j < blockDim.x; ++j) {
            block_sum += shared_mean_delta[j];
        }
        mean_delta[i] = block_sum * (-1.f / sqrtf(variance[i] + .00001f));
    }
}