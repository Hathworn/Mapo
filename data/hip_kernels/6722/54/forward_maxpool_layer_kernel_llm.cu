#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    // Compute dimensions for the output
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;

    // Calculate global index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Determine locations in output tensor
    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    // Offset values for padding
    int w_offset = -pad;
    int h_offset = -pad;

    // Calculate output index
    int out_index = j + w * (i + h * (k + c * b));
    
    // Initialize max value to negative infinity
    float max = -INFINITY;
    int max_i = -1;

    // Iterate over window to find maximum value
    for (int l = 0; l < size; ++l) {
        for (int m = 0; m < size; ++m) {
            int cur_h = h_offset + i * stride + l;
            int cur_w = w_offset + j * stride + m;
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));

            // Check if current position is within input bounds
            if (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w) {
                float val = input[index];
                if (val > max) {
                    max = val;
                    max_i = index;
                }
            }
        }
    }

    // Assign results to output and indexes
    output[out_index] = max;
    indexes[out_index] = max_i;
}