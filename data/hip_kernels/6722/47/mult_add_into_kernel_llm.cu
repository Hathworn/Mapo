#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Optimize index calculation for readability and performance
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize memory access pattern by using cooperative work size
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x) {
        c[idx] += a[idx] * b[idx];
    }
}