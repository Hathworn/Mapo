#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Pre-calculate the scaling factor for efficiency
    float scale = -.5f * powf(variance[i] + .00001f, (float)(-3.f/2.f));
    float sum = 0.0f;

    // Unroll the loop to improve performance
    for(int j = 0; j < batch; ++j){
        for(int k = 0; k < spatial; ++k){
            int index = j * filters * spatial + i * spatial + k;
            sum += delta[index] * (x[index] - mean[i]);
        }
    }

    // Store final result
    variance_delta[i] = sum * scale;
}