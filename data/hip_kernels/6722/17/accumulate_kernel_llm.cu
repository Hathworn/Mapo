#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use a 1D grid for simpler index computation
    if (i >= groups) return;
    
    float local_sum = 0; // Use a local variable to reduce global memory access
    for (int k = 0; k < n; ++k) {
        local_sum += x[k * groups + i];
    }
    sum[i] = local_sum; // Write back to global memory once
}