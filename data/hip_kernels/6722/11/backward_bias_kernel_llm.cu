#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK 1024 // Define BLOCK size

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Perform computation to reduce global memory accesses
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) {
            sum += delta[i + size * (filter + n * b)];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Reduce within shared memory for final result
    if (BLOCK >= 1024 && p < 512) { part[p] += part[p + 512]; } __syncthreads();
    if (BLOCK >= 512 && p < 256) { part[p] += part[p + 256]; } __syncthreads();
    if (BLOCK >= 256 && p < 128) { part[p] += part[p + 128]; } __syncthreads();
    if (BLOCK >= 128 && p < 64) { part[p] += part[p + 64]; } __syncthreads();
    if (p < 32) {
        volatile float* vpart = part; // Use volatile to prevent compiler optimizations
        vpart[p] += vpart[p + 32]; 
        vpart[p] += vpart[p + 16]; 
        vpart[p] += vpart[p + 8]; 
        vpart[p] += vpart[p + 4];
        vpart[p] += vpart[p + 2];
        vpart[p] += vpart[p + 1];
    }

    if (p == 0) {
        bias_updates[filter] += part[0];
    }
}