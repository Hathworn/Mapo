#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Calculate the index and feature pairs
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    
    // Return early for invalid pairs
    if (f2 <= f1) return;
    
    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;
    
    // Accumulate sums and norms
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            sum += output[i1] * output[i2];
            norm1 += output[i1] * output[i1];
            norm2 += output[i2] * output[i2];
        }
    }
    
    // Compute norms
    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    
    // Normalize sum
    sum /= norm;

    // Update delta array
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            float deltaFactor = -scale * sum / norm;
            delta[i1] += deltaFactor * output[i2];
            delta[i2] += deltaFactor * output[i1];
        }
    }
}