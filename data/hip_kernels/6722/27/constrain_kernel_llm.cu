#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Enable coalesced memory access by striding through INCX
    int index = i * INCX;

    // Ensure the thread processes an element within bounds
    if(index < N) {
        X[index] = fminf(ALPHA, fmaxf(-ALPHA, X[index]));
    }
}