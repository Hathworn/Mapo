#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x; // Optimize indexing using only blockIdx.x
    if (f >= n) return;

    float mean = 0.0f;
    for(int i = 0; i < size; ++i){
        mean += fabsf(weights[f*size + i]);
    }

    mean /= size; // Combined division operation
    for(int i = 0; i < size; ++i){
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
    }
}