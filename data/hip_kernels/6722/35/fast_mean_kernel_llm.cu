#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Use a single loop to reduce index computation overhead
    int total_spatial = spatial * batch;
    for (int index = id; index < total_spatial; index += threads) {
        int j = index / spatial;
        int i = index % spatial;
        int offset = j * spatial * filters + filter * spatial + i;
        local[id] += x[offset];
    }

    __syncthreads();

    // Use parallel reduction for sum
    if (threads >= 1024) { if (id < 512) { local[id] += local[id + 512]; } __syncthreads(); }
    if (threads >= 512)  { if (id < 256) { local[id] += local[id + 256]; } __syncthreads(); }
    if (threads >= 256)  { if (id < 128) { local[id] += local[id + 128]; } __syncthreads(); }
    if (threads >= 128)  { if (id < 64)  { local[id] += local[id + 64];  } __syncthreads(); }
    if (id < 32) {
        if (threads >= 64) local[id] += local[id + 32];
        local[id] += local[id + 16];
        local[id] += local[id + 8];
        local[id] += local[id + 4];
        local[id] += local[id + 2];
        local[id] += local[id + 1];
    }

    if (id == 0) {
        mean[filter] = local[0] / total_spatial;
    }
}