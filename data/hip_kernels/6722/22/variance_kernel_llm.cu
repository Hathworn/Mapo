#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Precompute reciprocal scale to avoid division in the loop
    float scale = 1.f / (batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float var_accumulator = 0.0f; // Use a register for accumulation

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            float diff = x[index] - mean[i];
            var_accumulator += diff * diff; // Avoid powf for better performance
        }
    }
    variance[i] = var_accumulator * scale; // Multiply scale at the end
}