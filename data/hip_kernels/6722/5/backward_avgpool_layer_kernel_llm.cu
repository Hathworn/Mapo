#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    int b = id / c;

    int out_index = k + c * b;
    float out_delta_val = out_delta[out_index] / (w * h); // Pre-compute division to avoid redundant calculations

    for(int i = 0; i < w * h; ++i){
        int in_index = i + h * w * (k + b * c);
        atomicAdd(&in_delta[in_index], out_delta_val); // Use atomicAdd for safe concurrent writes
    }
}