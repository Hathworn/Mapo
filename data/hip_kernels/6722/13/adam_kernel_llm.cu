#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplify grid-stride loop
    if (index < N) { // Change condition to a loop
        // Pre-compute constants outside the loop
        float B1_t = 1.f - powf(B1, t);
        float B2_t = 1.f - powf(B2, t);
        float lr_t = rate * sqrtf(B2_t) / B1_t;

        // Update variable
        x[index] += lr_t * m[index] / (sqrtf(v[index]) + eps);
    }
}