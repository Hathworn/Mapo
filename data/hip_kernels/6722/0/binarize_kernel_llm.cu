#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Use blockDim.x to maximize parallel threads
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for potential loop iteration
    while (i < n) {
        binary[i] = (x[i] >= 0) ? 1 : -1;
        i += stride; // Increment index by stride for loop iteration
    }
}