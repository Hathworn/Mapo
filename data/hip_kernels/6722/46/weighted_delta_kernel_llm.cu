#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Ensure the index is within bounds
    if(i < n){
        float dc_val = dc[i]; // Cache repeated memory fetch
        float s_val = s[i];   // Cache repeated memory fetch
        float a_val = a[i];   // Cache repeated memory fetch
        float b_val = b[i];   // Cache repeated memory fetch

        // Perform conditional updates
        if(da) da[i] += dc_val * s_val;
        if(db) db[i] += dc_val * (1.0f - s_val);
        ds[i] += dc_val * (a_val - b_val);
    }
}