#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return;

    // Pre-calculation of common terms to reduce operations
    int sample_w2 = w2 * sample;
    int sample_h2 = h2 * sample;
    int stride_w1 = w1 * stride;
    int stride_h1 = h1 * stride;

    int i = id % minw;
    int j = (id / minw) % minh;
    int k = (id / (minw * minh)) % minc;
    int b = id / (minw * minh * minc);

    int out_index = i * sample + sample_w2 * j + sample_h2 * (k + c2 * b);
    int add_index = i * stride + stride_w1 * j + stride_h1 * (k + c1 * b);

    out[out_index] += add[add_index];
}