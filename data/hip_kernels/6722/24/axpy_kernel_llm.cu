#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Use blockDim.x * gridDim.x for efficient grid-stride looping
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}