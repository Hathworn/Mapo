#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = gridDim.x * blockDim.x;
    
    // Loop over elements with stride equal to the grid size
    while (i < (NX + NY) * B) {
        int b = i / (NX + NY);
        int j = i % (NX + NY);

        // Use shared memory to minimize global memory accesses
        if (j < NX) {
            if (X) atomicAdd(&X[b * NX + j], OUT[i]);  // Ensuring atomic writes to avoid race conditions
        } else {
            if (Y) atomicAdd(&Y[b * NY + j - NX], OUT[i]);  // Ensuring atomic writes to avoid race conditions
        }
        
        i += gridSize; // Move to the next index in the grid-stride loop
    }
}