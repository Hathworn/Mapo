#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify index calculation
    if (i >= filters) return;

    float sum = 0.0f;  // Use a local variable for accumulation
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += x[index];  // Accumulate directly into local variable
        }
    }
    mean[i] = sum * scale;  // Store the scaled result
}