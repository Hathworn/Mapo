#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int batchSize = n * size;  // Precompute batch size

    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Simplified index calculation and conditional check
    if (offset < size) 
    {
        int idx = batch * batchSize + filter * size + offset;
        output[idx] *= biases[filter];
    }
}