#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK 256

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Optimize parallel accumulation across threads
    for(int b = 0; b < batch; ++b){
        for(int i = p; i < size; i += BLOCK){
            int index = i + size*(filter + n*b);
            sum += delta[index] * x_norm[index];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Optimize reduction using shared memory
    if (p < BLOCK/2) {
        part[p] += part[p + BLOCK/2];
    }
    __syncthreads();

    if (p < BLOCK/4) {
        part[p] += part[p + BLOCK/4];
    }
    __syncthreads();

    if (p < BLOCK/8) {
        part[p] += part[p + BLOCK/8];
    }
    __syncthreads();

    if (p < BLOCK/16) {
        part[p] += part[p + BLOCK/16];
    }
    __syncthreads();

    if (p == 0) {
        atomicAdd(&scale_updates[filter], part[0]);
    }
}