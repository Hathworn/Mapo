#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds and stride through data
    for (int i = idx; i < N; i += blockDim.x * gridDim.x)
    {
        Y[i * INCY] *= X[i * INCX];
    }
}