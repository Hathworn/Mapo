#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Initialize shared memory
    local[id] = 0.0f;

    // Loop over batch and spatial dimensions
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) { // Optimized loop to reduce divergence
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += powf((x[index] - mean[filter]), 2);
        }
    }

    __syncthreads();

    // Reduction in shared memory
    if (id == 0) {
        float sum = 0.0f;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        variance[filter] = sum / (spatial * batch - 1);
    }
}