#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate unique global thread index
    int stride = blockDim.x * gridDim.x;  // Calculate stride for loop

    for (int idx = index; idx < n * size * batch; idx += stride) {  // Loop over elements with stride
        int i = idx % size;  // Calculate local indices
        int j = (idx / size) % n;
        int k = (idx / (size * n));
        output[(k * n + j) * size + i] += biases[j];  // Perform bias addition
    }
}