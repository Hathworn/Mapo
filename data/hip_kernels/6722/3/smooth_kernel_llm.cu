#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int out_index = j + w * (i + h * (k + c * b));
    int w_offset = -(size / 2);
    int h_offset = -(size / 2);

    // Use shared memory to optimize access
    extern __shared__ float shared_x[];
    int shared_width = size + blockDim.x;
    int shared_height = size;

    // Load data into shared memory, handle boundary conditions
    if ((i + h_offset >= 0) && (i + h_offset < h) && (j + w_offset >= 0) && (j + w_offset < w)) {
        shared_x[threadIdx.x] = x[out_index];
    }

    __syncthreads();

    int l, m;
    float sum_delta = 0.0f;

    for (l = 0; l < size; ++l) {
        for (m = 0; m < size; ++m) {
            int cur_h = h_offset + i + l;
            int cur_w = w_offset + j + m;
            int index = cur_w + w * (cur_h + h * (k + b * c));

            if ((cur_h >= 0) && (cur_h < h) && (cur_w >= 0) && (cur_w < w)) {
                sum_delta += rate * (x[index] - x[out_index]);
            }
        }
    }

    delta[out_index] += sum_delta;
}