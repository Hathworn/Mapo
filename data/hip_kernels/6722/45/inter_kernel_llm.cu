#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    int stride = gridDim.x * blockDim.x; // Calculate the total stride

    while (i < (NX + NY) * B) { // Use loop to handle multiple elements per thread
        int b = i / (NX + NY);
        int j = i % (NX + NY);
        
        if (j < NX) {
            OUT[i] = X[b * NX + j];
        } else {
            OUT[i] = Y[b * NY + j - NX];
        }

        i += stride; // Move to the next element handled by this thread
    }
}