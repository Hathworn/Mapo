#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    float sum = 0.0f;
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) { // Use thread offset to balance load
            int index = j * spatial * filters + filter * spatial + i;
            sum += delta[index];
        }
    }
    local[id] = sum;

    __syncthreads();

    // Parallel reduction for local sum
    for (int offset = threads / 2; offset > 0; offset >>= 1) {
        if (id < offset) {
            local[id] += local[id + offset];
        }
        __syncthreads();
    }

    if (id == 0) {
        mean_delta[filter] = local[0] * (-1.f / sqrtf(variance[filter] + .00001f));
    }
}