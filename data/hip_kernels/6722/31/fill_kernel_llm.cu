#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Optimize by using stride loop
    for (int index = i; index < N; index += stride) {
        X[index * INCX] = ALPHA;
    }
}