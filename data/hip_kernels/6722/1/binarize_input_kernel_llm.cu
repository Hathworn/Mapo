#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;

    // Use shared memory to reduce global memory access
    extern __shared__ float shared_data[];
    float *shared_input = shared_data;
    float mean = 0;

    // Load input elements to shared memory
    for (int i = 0; i < n; ++i) {
        shared_input[i] = fabsf(input[i * size + s]);
    }
    __syncthreads();  // Ensure all data is loaded

    // Calculate mean
    for (int i = 0; i < n; ++i) {
        mean += shared_input[i];
    }
    mean = mean / n;

    // Binarize input
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}