#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation for 1D grid
    if (i < n) {
        float b_val = b ? b[i] : 0.0f; // Use local variable for conditional
        c[i] = s[i] * a[i] + (1 - s[i]) * b_val; // Simplified expression for readability
    }
}