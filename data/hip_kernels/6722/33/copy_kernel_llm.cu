#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * gridDim.x * blockDim.x);

    // Ensure the thread index is within bounds
    if(i < N) {
        // Perform the copy operation
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}