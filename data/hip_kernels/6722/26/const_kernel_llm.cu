#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Check if the index is within bounds and apply ALPHA
    if(i < N && INCX > 0) {
        X[i * INCX] = ALPHA;
    }
}