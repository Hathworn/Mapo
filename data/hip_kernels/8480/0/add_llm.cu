#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c) {
    // Use device memory to avoid bank conflicts and improve memory throughput
    __shared__ int temp;
    temp = a + b;
    // Ensure all threads have finished computing
    __syncthreads();
    if (threadIdx.x == 0) {
        *c = temp;
    }
}