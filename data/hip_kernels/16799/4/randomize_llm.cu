#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned char value(float n1, float n2, int hue) {
    if (hue > 360) hue -= 360;
    else if (hue < 0) hue += 360;

    if (hue < 60)
        return (unsigned char) (255 * (n1 + (n2 - n1) * hue / 60));
    if (hue < 180)
        return (unsigned char) (255 * n2);
    if (hue < 240)
        return (unsigned char) (255 * (n1 + (n2 - n1) * (240 - hue) / 60));
    return (unsigned char) (255 * n1);
}

__global__ void randomize(float* array, hiprandState* rand, unsigned long N) {
    // Compute tid using 2D grid and block dimensions
    unsigned long tid = threadIdx.x + blockIdx.x * blockDim.x + 
                        (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;

    if (tid < N) {
        // Use local hiprandState for random number generation
        hiprandState localState = rand[tid];
        float theRand = hiprand_uniform(&localState);
        
        // Save the modified state back for future usage
        rand[tid] = localState;

        // Assign the generated random number to the array
        array[tid] = theRand;
    }
}