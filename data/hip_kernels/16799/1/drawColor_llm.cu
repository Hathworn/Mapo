#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*******************************************************************************
*
*******************************************************************************/

/*************************************************************************

/*************************************************************************/

/*************************************************************************/
__global__ void drawColor(unsigned char* optr, const float* red, const float* green, const float* blue) {
    // Map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // Early exit if offset exceeds output size
    if (offset >= blockDim.x * gridDim.x * blockDim.y * gridDim.y) return;

    // Clamp and convert color components
    float theRed = fminf(fmaxf(red[offset], 0.0f), 1.0f);
    float theGreen = fminf(fmaxf(green[offset], 0.0f), 1.0f);
    float theBlue = fminf(fmaxf(blue[offset], 0.0f), 1.0f);

    // Write output color values
    optr[offset * 4 + 0] = static_cast<unsigned char>(255 * theRed);    // red
    optr[offset * 4 + 1] = static_cast<unsigned char>(255 * theGreen);  // green
    optr[offset * 4 + 2] = static_cast<unsigned char>(255 * theBlue);   // blue
    optr[offset * 4 + 3] = 255;                                         // alpha (opacity)
}