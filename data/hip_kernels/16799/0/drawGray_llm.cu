#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*******************************************************************************
*
*******************************************************************************/

/*************************************************************************

/*************************************************************************/

/*************************************************************************/
__global__ void drawGray(unsigned char* optr, const float* outSrc) {
    // Use block and thread indices to compute the pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // Load and normalize the value
    float val = outSrc[offset];
    val = fminf(fmaxf((val / 50.0f) + 0.5f, 0.0f), 1.0f);  // Clamp the normalized value

    // Pack the RGB values using a single operation
    unsigned char intensity = static_cast<unsigned char>(255 * val);
    int pixelOffset = offset * 4;
    optr[pixelOffset] = intensity;     // red
    optr[pixelOffset + 1] = intensity; // green
    optr[pixelOffset + 2] = intensity; // blue
    optr[pixelOffset + 3] = 255;       // alpha
}