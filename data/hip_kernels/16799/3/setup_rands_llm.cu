#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_rands(hiprandState* rand, unsigned long seed, unsigned long N)
{
    // Calculating unique thread index in the grid
    unsigned long tid = blockIdx.x * blockDim.x + threadIdx.x + 
                        (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x;

    // Ensure valid threads access
    if (tid < N) 
    {
        hiprand_init(seed, tid, 0, &rand[tid]);
    }
}