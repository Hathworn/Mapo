#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float* red, float* green, float* blue, unsigned long N){

    unsigned long tid = blockIdx.x * blockDim.x + threadIdx.x + 
                        (blockIdx.y * blockDim.y + threadIdx.y) * blockDim.x * gridDim.x;

    // Check tid and set values if within bounds
    if(tid < N){
        red[tid] = 0.5f;
        green[tid] = 0.2f;
        blue[tid] = 0.7f;
    }
}