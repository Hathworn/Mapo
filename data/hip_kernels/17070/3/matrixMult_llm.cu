#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMult(int* m, int* n, int* p, int size)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int p_sum = 0; // Initialize p_sum to 0

    if (row < size && col < size) { // Ensure thread is within bounds
        for (int i = 0; i < size; i++) {
            p_sum += m[row * size + i] * n[i * size + col]; // Correct index for n
        }
        p[row * size + col] = p_sum;
    }
}