#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMult(int* m, int* n, int* p, int size)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int p_sum = 0;  // Initialize sum to zero

    for (int i = 0; i < size; i++) {
        // Use coalesced memory access for m and n
        p_sum += m[row * size + i] * n[i * size + col];
    }
    
    p[row * size + col] = p_sum;
}