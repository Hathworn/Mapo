#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixSum(int* a, int* b, int* c, int size)
{
    // Calculate unique global thread ID
    int pos = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Globally synchronize using grid-stride loop
    for (int i = pos; i < size * size; i += blockDim.x * gridDim.x)
    {
        c[i] = a[i] + b[i];
    }
}