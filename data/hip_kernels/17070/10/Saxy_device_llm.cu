#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Saxy_device(float* x, float* y, float* d, float xb, float yb, int n)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (i < n)
    {
        // Perform vector operation
        d[i] = (x[i] - xb) * (y[i] - yb);
    }
}