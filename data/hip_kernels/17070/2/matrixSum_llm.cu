#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixSum(int* a, int* b, int* c, int size)
{
    int pos = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread position
    if (pos < size * size) {
        c[pos] = a[pos] + b[pos];
    }
}