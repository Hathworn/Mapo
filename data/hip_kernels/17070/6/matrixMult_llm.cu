#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMult(int* m, int* n, int* p, int size)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int p_sum = 0; // Initialize the sum to zero

    // Unroll loop for performance
    for (int i = 0; i < size; i++) {
        p_sum += m[row * size + i] * n[i * size + col]; // Fix index to correctly access n
    }

    if (row < size && col < size) { // Ensure within matrix bounds
        p[row * size + col] = p_sum;
    }
}