#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMult(int* m, int* n, int* p, int size)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize partial sum to zero
    int p_sum = 0;

    if(row < size && col < size) { // Check bounds to avoid invalid memory access
        for (int i = 0; i < size; i++) {
            // Accumulate product for current row and column
            p_sum += m[row * size + i] * n[i * size + col];
        }
        // Write result to output matrix
        p[row * size + col] = p_sum;
    }
}