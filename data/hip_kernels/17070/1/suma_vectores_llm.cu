#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void suma_vectores(int* c, const int* a, const int* b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Load data from global memory into register
    if (i < size) {
        int temp_a = a[i];
        int temp_b = b[i];
        // Perform computation using registers
        c[i] = temp_a + temp_b;
    }
}