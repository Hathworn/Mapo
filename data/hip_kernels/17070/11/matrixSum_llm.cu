#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixSum(int* a, int* b, int* c, int size)
{
    // Compute global thread position
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform addition if within bounds
    if (pos < size * size) {
        c[pos] = a[pos] + b[pos];
    }
}