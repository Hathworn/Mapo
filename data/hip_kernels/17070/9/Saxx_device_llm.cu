#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Saxx_device(float* x, float* c, float xb, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use block indexing for larger arrays
    if (i < n)
    {
        float diff = x[i] - xb; // Store result of subtraction
        c[i] = diff * diff;     // Square the difference
    }
}