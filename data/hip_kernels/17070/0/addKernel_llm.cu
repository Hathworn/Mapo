#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Helper function for using CUDA to add vectors in parallel.
__global__ void addKernel(int* c, const int* a, const int* b, int size) {
    // Use shared memory for coalesced memory access
    extern __shared__ int shared_a[];
    extern __shared__ int shared_b[];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Load data into shared memory
    if (i < size) {
        shared_a[tid] = a[i];
        shared_b[tid] = b[i];
    }
    __syncthreads();  // Synchronize threads

    // Perform vector addition
    if (i < size) {
        c[i] = shared_a[tid] + shared_b[tid];
    }
}