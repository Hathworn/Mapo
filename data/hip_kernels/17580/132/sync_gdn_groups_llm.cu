#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_gdn_groups() {
    // Use no-operation to synchronize threads as needed
    __syncthreads();  // Ensure all threads in the block reach this point
}