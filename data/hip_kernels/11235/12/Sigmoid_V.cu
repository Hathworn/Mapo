#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"






















__device__ hiprandState randomStates[256];



__global__ void Sigmoid_V(const float* a, float* out, const int n)
{
int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int i = blockId * blockDim.x + threadIdx.x;

if (i < n)
{
out[i] = 1.0f / (1.0f + __expf(-a[i]));
}
}