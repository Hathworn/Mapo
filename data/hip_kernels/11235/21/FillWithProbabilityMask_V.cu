#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"






















__device__ hiprandState randomStates[256];



__global__ void FillWithProbabilityMask_V(float* a, const float probability, int n)
{
int blockId = blockIdx.y * gridDim.x + blockIdx.x;
int i = blockId * blockDim.x + threadIdx.x;

if (i < n)
{
float rand = hiprand_uniform(&randomStates[i % 256]);

a[i] = rand < probability ? 1 : 0;
}
}