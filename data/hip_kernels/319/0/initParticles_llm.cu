#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
/// Copyright (C) 2016 Giuseppe Bilotta <giuseppe.bilotta@gmail.com>
/// License: GPLv3

#define restrict __restrict__

typedef unsigned int uint;
typedef unsigned int hashKey;
typedef ushort4 particleinfo;

__global__ void initParticles( particleinfo * restrict infoArray, hashKey * restrict hashArray, uint * restrict idxArray, uint numParticles)
{
    uint idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure idx is within bounds
    if (idx >= numParticles)
        return;

    // Directly assign index
    idxArray[idx] = idx;

    // Initialize particleinfo
    particleinfo info;
    info.x = idx & 3; // x optimized using bitwise AND for modulo 4
    info.y = 0;
    info.z = static_cast<ushort>(idx); // Use static_cast for type safety
    info.w = static_cast<ushort>(idx >> 16);

    infoArray[idx] = info;

    // Simplified hash calculation
    hashArray[idx] = idx / 17 + (idx % 17);
}