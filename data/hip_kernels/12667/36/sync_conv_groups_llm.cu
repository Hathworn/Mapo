#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize using __syncthreads() to ensure all threads in the block reach this point
__global__ void sync_conv_groups() { 
    __syncthreads(); // Global synchronization for the group
}