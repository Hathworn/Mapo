#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void updateCenters(float *centers, float *images, int *updates, int noClusters)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int imageSize = 784;
    int imagesOffset = gid * imageSize;

    float minValue = FLT_MAX; // Use FLT_MAX for better clarity
    int minCenterIndex = -1;
    
    // Load image data into shared memory to optimize access speed
    __shared__ float pImage[784]; 
    if (threadIdx.x < imageSize) {
        pImage[threadIdx.x] = images[imagesOffset + threadIdx.x];
    }
    __syncthreads();

    for (int centersIndex = 0; centersIndex < noClusters; centersIndex++)
    {
        float sum = 0.0f;
        for (int index = 0; index < imageSize; index++)
        {
            // Calculate squared distance
            float diff = centers[centersIndex * imageSize + index] - pImage[index];
            sum += diff * diff;
        }
        // Find the minimum squared distance
        if (sum < minValue)
        {
            minValue = sum;
            minCenterIndex = centersIndex;
        }
    }
    updates[gid] = minCenterIndex;
}