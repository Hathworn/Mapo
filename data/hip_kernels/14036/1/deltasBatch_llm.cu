#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"
__global__ void deltasBatch(float *inputs, float *outputs, float *weights, float *weightsDeltas, int noInputs, int inputSize) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    int offsetDeltas = (inputSize + 1) * gid;
    int offsetInput = noInputs * inputSize * gid;
    int offsetOutputs = noInputs * gid;

    // Initialize weightsDeltas to zero using one thread per element
    for (int imageIndex = threadIdx.x; imageIndex <= inputSize; imageIndex += blockDim.x) {
        weightsDeltas[offsetDeltas + imageIndex] = 0;
    }
    __syncthreads(); // Ensure all initializations are complete

    for (int i = 0; i < noInputs; i++) {
        sum = 0;
        // Optimize weighted sum calculation by using a single loop
        for (int imageIndex = 0; imageIndex < inputSize; imageIndex++) {
            sum += inputs[offsetInput + i * inputSize + imageIndex] * weights[imageIndex];
        }
        sum += weights[inputSize];
        sum = (sum > 0) ? 1 : 0; // Simplified ternary operation
        sum = outputs[offsetOutputs + i] - sum;

        if (sum != 0) {
            // Update weightsDeltas using strided access pattern
            for (int imageIndex = threadIdx.x; imageIndex < inputSize; imageIndex += blockDim.x) {
                atomicAdd(&weightsDeltas[offsetDeltas + imageIndex], sum * inputs[offsetInput + i * inputSize + imageIndex]);
            }
            if (threadIdx.x == 0) {
                atomicAdd(&weightsDeltas[offsetDeltas + inputSize], sum);
            }
        }
    }
}