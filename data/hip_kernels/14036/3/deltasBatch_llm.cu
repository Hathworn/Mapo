#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NO_HIDDEN_NEURONS 5

extern "C"
__global__ void deltasBatch(float *inputs, float *outputs, float *weights, float *weightsDeltas, int noInputs, int inputSize, int offsetHidden) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    int offsetDeltas = ((inputSize + 1) * NO_HIDDEN_NEURONS + NO_HIDDEN_NEURONS + 1) * gid;
    int offsetInput = noInputs * inputSize * gid;
    int offsetOutputs = noInputs * gid;
    float activationHidden[NO_HIDDEN_NEURONS];
    float error;
    int currentHidden = offsetHidden % NO_HIDDEN_NEURONS;
    
    // Initialize weightsDeltas to zero
    for (int imageIndex = 0; imageIndex <= inputSize; imageIndex++) {
        weightsDeltas[offsetDeltas + (inputSize + 1) * currentHidden + imageIndex] = 0;
    }
    weightsDeltas[offsetDeltas + (inputSize + 1) * NO_HIDDEN_NEURONS + currentHidden] = 0;

    for (int i = 0; i < noInputs; i++) {
        // Compute activation for hidden neurons
        for (int hidden = 0; hidden < NO_HIDDEN_NEURONS; hidden++) {
            sum = 0;
            for (int imageIndex = 0; imageIndex < inputSize; imageIndex++) {
                sum += inputs[offsetInput + i * inputSize + imageIndex] * weights[(inputSize + 1) * hidden + imageIndex];
            }
            sum += weights[(inputSize + 1) * hidden + inputSize];
            activationHidden[hidden] = (sum > 0) ? 1 : 0; // Apply ReLU activation
        }
        
        // Compute sum for output layer
        sum = 0;
        for (int hidden = 0; hidden < NO_HIDDEN_NEURONS; hidden++) {
            sum += activationHidden[hidden] * weights[(inputSize + 1) * NO_HIDDEN_NEURONS + hidden];
        }
        sum += weights[(inputSize + 1) * NO_HIDDEN_NEURONS + NO_HIDDEN_NEURONS];
        sum = (sum > 0) ? 1 : 0; // Apply step activation

        // Compute delta
        sum = outputs[offsetOutputs + i] - sum;
        if (sum != 0) {
            for (int hidden = 0; hidden < NO_HIDDEN_NEURONS; hidden++) {
                weightsDeltas[offsetDeltas + (inputSize + 1) * NO_HIDDEN_NEURONS + hidden] += sum * activationHidden[hidden];
            }
            weightsDeltas[offsetDeltas + (inputSize + 1) * NO_HIDDEN_NEURONS + NO_HIDDEN_NEURONS] += sum;

            error = sum * weights[(inputSize + 1) * NO_HIDDEN_NEURONS + currentHidden];
            error = (error > 0) ? 1 : 0; // Apply step function
            
            error -= activationHidden[currentHidden];
            if (error != 0) {
                for (int imageIndex = 0; imageIndex < inputSize; imageIndex++) {
                    weightsDeltas[offsetDeltas + (inputSize + 1) * currentHidden + imageIndex] += error * inputs[offsetInput + i * inputSize + imageIndex];
                }
                weightsDeltas[offsetDeltas + (inputSize + 1) * currentHidden + inputSize] += error;
            }
        }
    }
}