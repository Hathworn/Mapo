#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NO_HIDDEN_NEURONS 5

extern "C"
__global__ void deltasBatch(float *inputs, float *outputs, float *weights, float *weightsDeltas, int noInputs, int inputSize) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;
    int offsetDeltas = ((inputSize + 1) * NO_HIDDEN_NEURONS + NO_HIDDEN_NEURONS + 1) * gid;
    int offsetInput = noInputs * inputSize * gid;
    int offsetOutputs = noInputs * gid;
    float activationHidden[NO_HIDDEN_NEURONS];
    float error;

    // Efficiently zero out weightsDeltas
    for (int j = 0; j < (inputSize + 1) * (NO_HIDDEN_NEURONS + 1); j++) {
        weightsDeltas[offsetDeltas + j] = 0;
    }

    for (int i = 0; i < noInputs; i++) {
        // Vectorized computation for hidden layer activations
        for (int hidden = 0; hidden < NO_HIDDEN_NEURONS; hidden++) {
            sum = weights[(inputSize + 1) * hidden + inputSize]; // Bias term
            for (int imageIndex = 0; imageIndex < inputSize; imageIndex++) {
                sum += inputs[offsetInput + i * inputSize + imageIndex] * weights[(inputSize + 1) * hidden + imageIndex];
            }
            activationHidden[hidden] = (sum > 0) ? 1 : 0;
        }

        // Compute output layer sum
        sum = weights[(inputSize + 1) * NO_HIDDEN_NEURONS + NO_HIDDEN_NEURONS]; // Bias term
        for (int hidden = 0; hidden < NO_HIDDEN_NEURONS; hidden++) {
            sum += activationHidden[hidden] * weights[(inputSize + 1) * NO_HIDDEN_NEURONS + hidden];
        }
        sum = (sum > 0) ? 1 : 0;
        sum = outputs[offsetOutputs + i] - sum;

        if (sum != 0) {
            // Update weightsDeltas for output layer
            for (int hidden = 0; hidden < NO_HIDDEN_NEURONS; hidden++) {
                weightsDeltas[offsetDeltas + (inputSize + 1) * NO_HIDDEN_NEURONS + hidden] += sum * activationHidden[hidden];
            }
            weightsDeltas[offsetDeltas + (inputSize + 1) * NO_HIDDEN_NEURONS + NO_HIDDEN_NEURONS] += sum;

            // Propagate error back to hidden layer
            for (int hidden = 0; hidden < NO_HIDDEN_NEURONS; hidden++) {
                error = (sum * weights[(inputSize + 1) * NO_HIDDEN_NEURONS + hidden] > 0) ? 1 : 0;
                error -= activationHidden[hidden];
                
                if (error != 0) {
                    // Efficiently update weightsDeltas for hidden layer
                    for (int imageIndex = 0; imageIndex < inputSize; imageIndex++) {
                        weightsDeltas[offsetDeltas + (inputSize + 1) * hidden + imageIndex] += error * inputs[offsetInput + i * inputSize + imageIndex];
                    }
                    weightsDeltas[offsetDeltas + (inputSize + 1) * hidden + inputSize] += error;
                }
            }
        }
    }
}