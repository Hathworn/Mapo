#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

extern "C"
__global__ void deltasOne(float *inputs, float *outputs, float *weights, float *weightsDeltas, int offsetInputImages, int inputSize) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= inputSize) return; // Early exit if gid is out of bounds
    
    float sum = 0;
    int offsetDeltas = (inputSize + 1) * gid;
    int offsetInput = inputSize * (gid + offsetInputImages);

    // Initialize weightsDeltas with a single call to memset
    for (int imageIndex = 0; imageIndex <= inputSize; imageIndex++) {
        weightsDeltas[offsetDeltas + imageIndex] = 0;
    }

    // Use a single loop and accumulate
    for (int imageIndex = 0; imageIndex < inputSize; imageIndex++) {
        sum += inputs[offsetInput + imageIndex] * weights[imageIndex];
    }
    sum += weights[inputSize];
    sum = outputs[offsetInputImages + gid] - (sum > 0 ? 1 : 0);

    if (sum != 0) {
        for (int imageIndex = 0; imageIndex < inputSize; imageIndex++) {
            weightsDeltas[offsetDeltas + imageIndex] += sum * inputs[offsetInput + imageIndex];
        }
        weightsDeltas[offsetDeltas + inputSize] += sum;
    }
}