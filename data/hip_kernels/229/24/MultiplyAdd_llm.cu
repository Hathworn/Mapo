#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Compute x and y using blockDim instead of fixed value for better flexibility
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int p = y * width + x; // Simplified index computation
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
    // No need for __syncthreads() as there are no shared memory conflicts
}