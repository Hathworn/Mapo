#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    // Use blockDim.x and blockDim.y to improve flexibility
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * pitch + x;

    // Remove unnecessary __syncthreads as there's no shared memory usage
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}