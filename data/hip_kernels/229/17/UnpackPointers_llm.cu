#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;
    // Unroll outer loop to improve memory coalescing
    for (int y = 0; y < h; y += 32) {
        for (int x = 0; x < w; x += 16) {
            unsigned int val = minmax[(y/32)*w + x + tx];
            if (val) {
                #pragma unroll // Unrolling inner loop for better performance
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}