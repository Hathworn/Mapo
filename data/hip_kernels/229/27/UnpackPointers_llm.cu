#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;

    // Optimize loop unrolling and memory access
    for (int y = 0; y < h / 32; y++) {
        for (int x = 0; x < w; x += 16) {
            unsigned int val = minmax[y * w + x + tx];
            int ptrIndex = 0;

            while (val && numPts < maxPts) {
                if (val & 0x1) {
                    ptrs[16 * numPts + tx] = (y * 32 + ptrIndex) * w + x + tx;
                    numPts++;
                }
                val >>= 1;
                ptrIndex++;
            }
        }
    }
}