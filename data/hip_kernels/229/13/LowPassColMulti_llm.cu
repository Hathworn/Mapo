#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassColMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVCOL_W*(CONVCOL_H + 2*RADIUS)];

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int block = blockIdx.x / (NUM_SCALES + 3);
    const int scale = blockIdx.x - (NUM_SCALES + 3) * block;
    const int miny = blockIdx.y * CONVCOL_H;
    const int maxy = min(miny + CONVCOL_H, height) - 1;
    const int totStart = miny - RADIUS;
    const int totEnd = maxy + RADIUS;
    const int colStart = block * CONVCOL_W + tx;
    const int colEnd = colStart + (height - 1) * pitch;
    const int sStep = CONVCOL_W * CONVCOL_S;
    const int gStep = pitch * CONVCOL_S;

    float *kernel = d_Kernel + scale * 16;
    const int size = pitch * height * scale;
    d_Result += size;
    d_Data += size;

    if (colStart < width) {
        float *sdata = data + ty * CONVCOL_W + tx;
        int gPos = colStart + (totStart + ty) * pitch;

        // Prefetch data to shared memory
        for (int y = totStart + ty; y <= totEnd; y += blockDim.y) {
            if (y < 0)
                sdata[0] = d_Data[colStart];
            else if (y >= height)
                sdata[0] = d_Data[colEnd];
            else
                sdata[0] = d_Data[gPos];

            sdata += sStep;
            gPos += gStep;
        }
    }
    __syncthreads();

    if (colStart < width) {
        float *sdata = data + ty * CONVCOL_W + tx;
        int gPos = colStart + (miny + ty) * pitch;

        // Perform convolution in shared memory
        for (int y = miny + ty; y <= maxy; y += blockDim.y) {
            float sum = 0.0f;
            #pragma unroll // Unroll loop for performance
            for (int k = 0; k <= 4; ++k) {
                sum += (sdata[k * CONVCOL_W] + sdata[(8 - k) * CONVCOL_W]) * kernel[k];
            }
            sum -= sdata[4 * CONVCOL_W] * kernel[4]; // Adjust center term

            d_Result[gPos] = sum;
            sdata += sStep;
            gPos += gStep;
        }
    }
}