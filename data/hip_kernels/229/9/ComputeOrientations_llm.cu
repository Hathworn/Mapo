#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOrientations(float *g_Data, float *d_Sift, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    
    if (tx < 13) {
        for (int i = 0; i < 32; i++) // Initialize histogram with zero
            hist[i * 13 + tx] = 0.0f;
    }
    __syncthreads();

    float i2sigma2 = -1.0f/(2.0f*3.0f*3.0f);
    if (tx < 15)
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));

    int xp = (int)(d_Sift[bx + 0*maxPts] - 6.5f);
    int yp = (int)(d_Sift[bx + 1*maxPts] - 6.5f);
    int px = xp & 15;
    int x = tx - px;

    for (int y = 0; y < 15; y++) {
        int memPos = 16*y + x;
        int xi = xp + x;
        int yi = yp + y;
        xi = min(max(xi, 0), w-1); // Bound check for xi
        yi = min(max(yi, 0), h-1); // Bound check for yi
        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = 16.0f * atan2f(dy, dx) / 3.1416f + 16.5f;
            bin = (bin == 32) ? 0 : bin; // Avoid overflow in binning
            float grad = sqrtf(dx * dx + dy * dy);
            atomicAdd(&hist[32 * (x - 1) + bin], grad * gauss[x] * gauss[y]); // Use atomicAdd for reduction
        }
    }
    __syncthreads();

    for (int offset = 8; offset > 0; offset >>= 1) { // Reduce summation steps
        if (tx < 32)
            hist[tx] += hist[tx + offset * 32];
        __syncthreads();
    }
    
    if (tx < 32) {
        float v = hist[32 + tx];
        hist[tx] = (v > hist[32 + ((tx + 1) & 31)] && v >= hist[32 + ((tx + 31) & 31)]) ? v : 0.0f;
    }
    __syncthreads();
    
    if (tx == 0) {
        float maxval1 = 0.0, maxval2 = 0.0, val1, val2, peak;
        int i1 = -1, i2 = -1;
        
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }
        
        val1 = hist[32 + ((i1 + 1) & 31)];
        val2 = hist[32 + ((i1 + 31) & 31)];
        peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Sift[bx + 5 * maxPts] = 11.25f * ((peak < 0.0f) ? peak + 32.0f : peak);

        if (maxval2 < 0.8f * maxval1)
            i2 = -1;

        if (i2 >= 0) {
            val1 = hist[32 + ((i2 + 1) & 31)];
            val2 = hist[32 + ((i2 + 31) & 31)];
            peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Sift[bx + 6 * maxPts] = 11.25f * ((peak < 0.0f) ? peak + 32.0f : peak);
        } else {
            d_Sift[bx + 6 * maxPts] = i2;
        }
    }
}