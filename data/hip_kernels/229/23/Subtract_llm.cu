#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Replace deprecated __mul24
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Replace deprecated __mul24
    int p = y * width + x; // Simplify expression
    if (x < width && y < height) // Check boundary conditions
    {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // __syncthreads() is unnecessary here as each thread updates distinct memory
}