#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    
    // Initialize histogram
    for (int i = 0; i < 13; i++) hist[i*32 + tx] = 0.0f;
    __syncthreads();

    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15) gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));
    
    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;
    int px = xp & 15;
    int x = tx - px;
    
    // Load data into shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = min(max(xp + x, 0), w - 1);  // Clamp index within valid range
        int yi = min(max(yp + y, 0), h - 1);
        if (x >= 0 && x < 15) data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Calculate gradients and populate histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = 16.0f * atan2f(dy, dx) / 3.1416f + 16.5f;
            if (bin == 32) bin = 0;
            float grad = sqrtf(dx * dx + dy * dy);
            atomicAdd(&hist[32 * (x - 1) + bin], grad * gauss[x] * gauss[y]); // Use atomicAdd to avoid race conditions
        }
    }
    __syncthreads();

    // Reduce histogram
    for (int offset = 8; offset > 0; offset >>= 1) {
        if (tx < offset) 
            hist[tx] += hist[tx + offset];
        __syncthreads();
    }

    // Smooth histogram
    if (tx < 32) {
        float temp = hist[tx];
        hist[tx + 32] = 6 * temp + 4 * (hist[((tx + 1) & 31) + 32] + hist[((tx + 31) & 31) + 32]) +
                        (hist[((tx + 2) & 31) + 32] + hist[((tx + 30) & 31) + 32]);
    }
    __syncthreads();

    float v = hist[32 + tx];
    hist[tx] = (v > hist[32 + ((tx + 1) & 31)] && v >= hist[32 + ((tx + 31) & 31)] ? v : 0.0f);
    __syncthreads();

    // Find peaks in histogram
    if (tx == 0) {
        float maxval1 = 0.0f, maxval2 = 0.0f;
        int i1 = -1, i2 = -1;
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }

        if (maxval1 > 0) {
            float val1 = hist[32 + ((i1 + 1) & 31)];
            float val2 = hist[32 + ((i1 + 31) & 31)];
            float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
            d_Orient[bx] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        }
        
        if (maxval2 < 0.8f * maxval1)
            i2 = -1;

        if (i2 >= 0) {
            float val1 = hist[32 + ((i2 + 1) & 31)];
            float val2 = hist[32 + ((i2 + 31) & 31)];
            float peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Orient[bx + maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        } else {
            d_Orient[bx + maxPts] = i2;
        }
    }
}