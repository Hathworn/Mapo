#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
    int i = blockIdx.x * POSBLK_SIZE + threadIdx.x; // Use simple arithmetic for index calculation
    if (i >= numPts)
        return;

    int p = d_Ptrs[i];

    // Pre-fetch data to reduce redundant global memory accesses
    float val0 = g_Data2[p];
    float val_m1 = g_Data2[p - 1];
    float val_p1 = g_Data2[p + 1];
    float val_mw = g_Data2[p - w];
    float val_pw = g_Data2[p + w];
    float val3 = g_Data3[p];
    float val1 = g_Data1[p];

    // Compute gradients
    float dx = 0.5f * (val_p1 - val_m1);
    float dxx = 2.0f * val0 - val_m1 - val_p1;
    float dy = 0.5f * (val_pw - val_mw);
    float dyy = 2.0f * val0 - val_mw - val_pw;
    float ds = 0.5f * (val1 - val3);
    float dss = 2.0f * val0 - val3 - val1;

    // Compute mixed derivatives
    float dxy = 0.25f * (g_Data2[p + w + 1] + g_Data2[p - w - 1] - g_Data2[p - w + 1] - g_Data2[p + w - 1]);
    float dxs = 0.25f * (g_Data3[p + 1] + g_Data1[p - 1] - g_Data1[p + 1] - g_Data3[p - 1]);
    float dys = 0.25f * (g_Data3[p + w] + g_Data1[p - w] - g_Data3[p - w] - g_Data1[p + w]);

    // Compute matrix determinant and inverse
    float idxx = dyy * dss - dys * dys;
    float idxy = dys * dxs - dxy * dss;
    float idxs = dxy * dys - dyy * dxs;
    float idyy = dxx * dss - dxs * dxs;
    float idys = dxy * dxs - dxx * dys;
    float idss = dxx * dyy - dxy * dxy;
    float det = idxx * dxx + idxy * dxy + idxs * dxs;
    float idet = 1.0f / det;

    // Compute displacements
    float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
    float pdy = idet * (idxy * dx + idyy * dy + idys * ds);
    float pds = idet * (idxs * dx + idys * dy + idss * ds);

    // Check bounds and adjust displacements
    if (pdx < -0.5f || pdx > 0.5f || pdy < -0.5f || pdy > 0.5f || pds < -0.5f || pds > 0.5f) {
        pdx = __fdividef(dx, dxx);
        pdy = __fdividef(dy, dyy);
        pds = __fdividef(ds, dss);
    }

    // Compute the final sift descriptor values
    float dval = 0.5f * (dx * pdx + dy * pdy + ds * pds);
    d_Sift[i + 0 * maxPts] = (p % w) + pdx;
    d_Sift[i + 1 * maxPts] = (p / w) + pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val0 + dval;
    float tra = dxx + dyy;
    det = dxx * dyy - dxy * dxy;
    d_Sift[i + 4 * maxPts] = __fdividef(tra * tra, det);
}