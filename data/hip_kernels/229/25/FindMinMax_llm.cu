#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx; // Simplified index calculation
    const int y = blockIdx.y * 16;
    const int blockSize = blockDim.x;
    int p = y * width + x;

    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        minvals[tx] = FLOAT_MAX; // Initialize to max possible float
        maxvals[tx] = -FLOAT_MAX; // Initialize to min possible float
    }

    // Accumulate min and max values for the block
    for (int ty = 1; ty < 16; ++ty) {
        p += width;
        if (x < width && y + ty < height) {
            float val = d_Data[p];
            if (val < minvals[tx]) minvals[tx] = val;
            if (val > maxvals[tx]) maxvals[tx] = val;
        }
    }
    __syncthreads();

    // Reduction using parallel tree
    for (int stride = blockSize / 2; stride > 0; stride >>= 1) {
        if (tx < stride) {
            if (minvals[tx + stride] < minvals[tx]) minvals[tx] = minvals[tx + stride];
            if (maxvals[tx + stride] > maxvals[tx]) maxvals[tx] = maxvals[tx + stride];
        }
        __syncthreads();
    }

    // Write the block's min and max to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}