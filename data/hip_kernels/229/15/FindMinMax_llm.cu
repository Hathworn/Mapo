#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * blockDim.x + tx; // Use blockDim.x instead of fixed value
    const int y = blockIdx.y * 16;
    int p = y * pitch + x;

    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
        for (int ty = 1; ty < 16; ty++) {
            p += pitch;
            val = d_Data[p];
            minvals[tx] = fminf(minvals[tx], val); // Use fminf for readability
            maxvals[tx] = fmaxf(maxvals[tx], val); // Use fmaxf for readability
        }
    } else {
        minvals[tx] = FLT_MAX;  // Avoid reading out of bounds, use FLT_MAX
        maxvals[tx] = FLT_MIN;  // Use FLT_MIN
    }
    __syncthreads();
    
    for (int d = blockDim.x / 2; d > 0; d >>= 1) { // Iterate from half blockDim.x downward
        if (tx < d && (tx + d) < blockDim.x) { // Avoid out of bounds
            minvals[tx] = fminf(minvals[tx], minvals[tx + d]); // Use fminf
            maxvals[tx] = fmaxf(maxvals[tx], maxvals[tx + d]); // Use fmaxf
        }
        __syncthreads();
    }
    
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}