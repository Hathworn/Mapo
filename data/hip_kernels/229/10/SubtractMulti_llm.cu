#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Calculate global thread indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate the size of a slice (optimized thread index calculation)
    int sz = height * pitch;
    int p = threadIdx.z * sz + y * pitch + x;

    // Ensure the computation is only performed within bounds
    if (x < width && y < height) {
        // Subtract the corresponding elements and store the result
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
}