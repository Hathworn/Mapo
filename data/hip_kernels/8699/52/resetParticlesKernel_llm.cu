#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void resetParticlesKernel(float3* pos, float3* vel, float* age, float* life, char* opafix, int nParts)
{
    // Use blockIdx.x * blockDim.x + threadIdx.x as the thread's global index
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread index is within the bounds
    if (x < nParts) {
        // Reset particle properties using single assignment and reduce redundant calculations
        pos[x] = make_float3(2.0f, 0.0f, 0.0f);
        vel[x] = make_float3(0.0f, 0.0f, 0.0f);
        age[x] = life[x] = -1.0f;
        opafix[x] = 0;
    }
}