#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use a shared memory location to perform division only once
    __shared__ float shared_accuracy;
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy / N;
    }
    __syncthreads();
    if (threadIdx.x == 0) {
        *accuracy = shared_accuracy;
    }
}