#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Optimize memory access by using shared memory state
    __shared__ hiprandState sharedState[NUM_RND_THREADS_PER_BLOCK];
    // Initialize RNG in shared memory
    hiprand_init(seed, tidx, 0, &sharedState[threadIdx.x]);
    // Copy shared RNG state to global memory
    state[tidx] = sharedState[threadIdx.x];
}