#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel( const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Optimize: Unroll loop for performance
    #pragma unroll
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Optimize: Use warp-level reduction for faster computation
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (idx < offset) {
            reduction_buffer[idx] += reduction_buffer[idx + offset];
        }
        __syncthreads();
    }

    // Compute gradient using the reduced value
    tmp = reduction_buffer[0];
    #pragma unroll
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}