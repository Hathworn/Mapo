#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

#define THREADS_PER_BLOCK 32
#define NUM_BLOCKS 32

typedef double HighlyPrecise;

const int GENOME_LENGTH = 14;
const int GENE_MAX = 1;

const float MUTATION_FACTOR = 0.2;
const float CROSSOVER_RATE = 0.6;

const int NUM_EPOCHS = 1000;

struct Chromosome {
    HighlyPrecise genes[GENOME_LENGTH];
    HighlyPrecise fitnessValue;
};

// Optimize global kernel
__global__ void setupRandomStream(unsigned int seed, hiprandState* states) {
    // Use grid-stride loop for better resource utilization
    int threadIndex = blockDim.x * blockIdx.x + threadIdx.x; 
    int stride = blockDim.x * gridDim.x;
    for (int i = threadIndex; i < NUM_BLOCKS * THREADS_PER_BLOCK; i += stride) {
        hiprand_init(seed, i, 0, &states[i]);
    }
}