#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**
 * Optimized Kronecker product of two matrices kernel
 * input :
 * a : first matrix
 * nax, nay : matrix a dimensions
 * b: second matrix
 * nbx, nby : matrix b dimensions
 * results : kronecker product of a and b
 **/

__global__ void pick(size_t N , size_t * d_dst, unsigned long seed)
{
    int myId = blockIdx.x * blockDim.x + threadIdx.x;
    if (myId >= N)
        return;

    hiprandState state;
    // Initialize hiprand state with unique per thread stride for efficiency
    hiprand_init(seed, myId, 0, &state);

    // Use volatile to prevent compiler optimizations influencing random number generation
    volatile float RANDOM = hiprand_uniform(&state);
    d_dst[myId] = (size_t)(RANDOM * N);  // Remove unnecessary subtraction for efficiency
}