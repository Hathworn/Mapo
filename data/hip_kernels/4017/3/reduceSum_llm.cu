#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized reduceSum kernel
__global__ void reduceSum(double* d_arr, const size_t sz, double* d_out) {
    extern __shared__ double sh_out[];
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tId = threadIdx.x;

    // Initialize shared memory to zero for out-of-bounds threads
    sh_out[tId] = (myId < sz) ? d_arr[myId] : 0.0;
    __syncthreads();

    // Reduce the shared memory using a binary tree reduction
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tId < s) {
            sh_out[tId] += sh_out[tId + s];
        }
        __syncthreads();
    }

    // Write result from the first thread
    if (tId == 0) {
        d_out[blockIdx.x] = sh_out[0];
    }
}