#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findID(double *a, int n) {
    int tPosX = blockIdx.x * blockDim.x + threadIdx.x;
    // Use less divergence by shifting out of the condition
    if (tPosX >= n) return; // Handle boundary condition at the start
    a[tPosX] = tPosX; // Compute global ID and assign to array
}