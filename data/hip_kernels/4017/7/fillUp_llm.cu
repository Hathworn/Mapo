#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**
* Kronecker product of two matrices kernel
* input :
* a : first matrix
* nax, nay : matrix a dimensions
* b: second matrix
* nbx, nby : matrix b dimensions
* results : kronecker product of a and b
**/

__global__ void fillUp(size_t * d_dst, size_t N) {
    int myId = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single conditional check to improve efficiency
    if (myId < N) {
        d_dst[myId] = myId;
    }
}