#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kronecker product of two matrices kernel

__global__ void randomKey(size_t N, float *d_dst, unsigned long seed) {
    // Calculate global thread index
    int myId = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only proceed if within bounds
    if (myId < N) {
        // Initialize random state with unique seed
        hiprandState state;
        hiprand_init(seed, myId, 0, &state);
        
        // Generate random number
        float randomVal = hiprand_uniform(&state);
        
        // Store random value in destination array
        d_dst[myId] = randomVal;
    }
}