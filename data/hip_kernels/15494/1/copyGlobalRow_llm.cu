#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyGlobalRow(float *in, float *out, const int nx, const int ny)
{
    // Calculate a single thread's global index
    unsigned int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    // Use a loop to process multiple elements per thread if necessary
    for (unsigned int globalPos = globalIndex; globalPos < nx * ny; globalPos += stride)
    {
        out[globalPos] = in[globalPos];
    }
}