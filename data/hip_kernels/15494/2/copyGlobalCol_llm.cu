#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyGlobalCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate flattened index directly for coalesced memory access
    unsigned int idx = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if (idx < nx && idy < ny)
    {
        unsigned int index = idx * ny + idy;
        out[index] = in[index];
    }
}