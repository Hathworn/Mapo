#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeGlobalCol(float *in, float *out, const int nx, const int ny)
{
    // Calculate global thread indices
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int j = threadIdx.y + blockDim.y * blockIdx.y;

    // Use shared memory to optimize memory access
    __shared__ float tile[32][32 + 1]; // Use padding to avoid bank conflicts 

    if (i < nx && j < ny)
    {
        // Load data from global to shared memory
        tile[threadIdx.y][threadIdx.x] = in[i * ny + j];

        // Synchronize to ensure all loads are complete
        __syncthreads();

        // Write transposed data from shared to global memory
        out[j * nx + i] = tile[threadIdx.y][threadIdx.x];
    }
}