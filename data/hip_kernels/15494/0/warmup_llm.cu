#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *in, float *out, const int nx, const int ny)
{
    // Cache indices to eliminate redundant operations
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int j = threadIdx.y + blockDim.y * blockIdx.y;
    
    // Use shared memory if possible for more efficient data access
    __shared__ float cache[32][32];  // Example size, adjust as needed for your kernel

    if (i < nx && j < ny)
    {
        // Cache the input value to reduce global memory traffic
        cache[threadIdx.y][threadIdx.x] = in[j * nx + i];
        __syncthreads();

        // Write the output value from shared memory
        out[j * nx + i] = cache[threadIdx.y][threadIdx.x];
    }
}