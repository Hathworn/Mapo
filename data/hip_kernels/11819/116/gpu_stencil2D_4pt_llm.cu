#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double *dst, double *src, int M, int N) {
    // Using external shared memory
    extern __shared__ double shared_mem[];
    double *shSrc = shared_mem;

    // Calculate thread indices and shared memory dimensions
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int blockStartIdxX = blockIdx.x * blockDim.x * TILE_SIZE;
    int blockStartIdxY = blockIdx.y * blockDim.y * TILE_SIZE;
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Calculate global and shared memory indices
    int globalBaseIndex = HALO * N + blockStartIdxX * N + blockStartIdxY + HALO;
    int shBaseIndex = HALO * smColDim + HALO;

    // Load inner tile into shared memory
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = globalBaseIndex + tx * TILE_SIZE * N + ty * TILE_SIZE + i * N + j;
            int shMemIndex = shBaseIndex + tx * smColDim * TILE_SIZE + ty * TILE_SIZE + i * smColDim + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Load halo regions into shared memory: executed by a single thread
    if (tx == 0 && ty == 0) {
        int index;
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smColDim; j++) {
                index = (blockStartIdxX + i) * N + blockStartIdxY + j;
                shSrc[i * smColDim + j] = src[index];
                index = (HALO + blockStartIdxX + blockDim.x * TILE_SIZE + i) * N + blockStartIdxY + j;
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[index];
            }
        }
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smRowDim - HALO * 2; j++) {
                index = (HALO + blockStartIdxX + j) * N + blockStartIdxY + i;
                shSrc[(HALO + j) * smColDim + i] = src[index];
                index = (HALO + blockStartIdxX + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[index];
            }
        }
    }

    __syncthreads();

    // Compute the stencil operation for each inner element
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = globalBaseIndex + tx * TILE_SIZE * N + ty * TILE_SIZE + i * N + j;
            int shMemIndex = shBaseIndex + tx * smColDim * TILE_SIZE + ty * TILE_SIZE + i * smColDim + j;
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }

    __syncthreads();
}