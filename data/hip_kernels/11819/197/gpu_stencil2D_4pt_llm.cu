#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized for memory access
__global__ void gpu_stencil2D_4pt(double *dst, double *src, int M, int N) {
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double *shSrc = shared_mem;

    // indexes
    int i, j;

    // neighbor's values
    double north, south, east, west;

    // SharedMem Columns Dimension
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Calculate global and shared indices
    int blockOffsetX = blockIdx.x * blockDim.x * TILE_SIZE;
    int blockOffsetY = blockIdx.y * blockDim.y * TILE_SIZE;
    int threadOffsetX = threadIdx.x * TILE_SIZE;
    int threadOffsetY = threadIdx.y * TILE_SIZE;
    int baseGlobalIndex = HALO * N + blockOffsetX * N + blockOffsetY;
    int baseShMemIndex = HALO * smColDim + threadOffsetX * smColDim + HALO + threadOffsetY;

    // Copying to shared memory (Inner part)
    for (i = 0; i < TILE_SIZE; i++) {
        for (j = 0; j < TILE_SIZE; j++) {
            int globalIndex = baseGlobalIndex + threadOffsetX * N + i * N + threadOffsetY + j;
            int shMemIndex = baseShMemIndex + i * smColDim + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Synchronize threads before halo data read
    __syncthreads();

    // Halos (optimized halo copying)
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;
        // For Bottom and top row
        for (i = 0; i < HALO; i++) {
            for (j = 0; j < smColDim; j++) {
                indexTopHalo = blockOffsetX * N + i * N + blockOffsetY + j;
                indexBottomHalo = (HALO + blockOffsetX + blockDim.x * TILE_SIZE) * N + blockOffsetY + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }
        // For right and left Columns
        for (i = 0; i < HALO; i++) {
            for (j = 0; j < smRowDim - HALO * 2; j++) {
                indexLeftHalo = (HALO + blockOffsetX + j) * N + blockOffsetY + i;
                indexRightHalo = (HALO + blockOffsetX + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    // Synchronize threads to ensure halo data is copied
    __syncthreads();

    // Compute the stencil operation
    for (i = 0; i < TILE_SIZE; i++) {
        for (j = 0; j < TILE_SIZE; j++) {
            int globalIndex = baseGlobalIndex + threadOffsetX * N + i * N + threadOffsetY + j;
            int shMemIndex = baseShMemIndex + i * smColDim + j;
            // Getting the neighbors
            north = shSrc[shMemIndex - smColDim];
            south = shSrc[shMemIndex + smColDim];
            east = shSrc[shMemIndex + 1];
            west = shSrc[shMemIndex - 1];
            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
    
    // Final synchronization
    __syncthreads();
}