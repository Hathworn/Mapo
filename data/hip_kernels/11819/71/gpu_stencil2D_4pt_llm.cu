#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N) {
  // Declaring the shared memory array for source
  extern __shared__ double shared_mem[];
  double * shSrc = shared_mem;

  // SharedMem Columns Dimension
  int smColDim = HALO*2 + blockDim.y*TILE_SIZE;
  int smRowDim = HALO*2 + blockDim.x*TILE_SIZE;

  // Calculate global indices for the thread
  int baseGlobalRow = blockIdx.x * blockDim.x * TILE_SIZE + threadIdx.x * TILE_SIZE + HALO;
  int baseGlobalCol = blockIdx.y * blockDim.y * TILE_SIZE + threadIdx.y * TILE_SIZE + HALO;

  // Copying to shared memory: Inner part
  for (int i = 0; i < TILE_SIZE; i++) {
    for (int j = 0; j < TILE_SIZE; j++) {
      int globalIndex = (baseGlobalRow + i) * N + (baseGlobalCol + j);
      int shMemIndex = (HALO + threadIdx.x * TILE_SIZE + i) * smColDim + (HALO + threadIdx.y * TILE_SIZE + j);
      shSrc[shMemIndex] = src[globalIndex];
    }
  }

  // Synchronize to ensure all threads have copied to shared memory
  __syncthreads();

  // Load Halo regions only with a single thread to minimize redundant loads
  if (threadIdx.x == 0 && threadIdx.y == 0) {
    int index;
    
    // Top and Bottom Halos
    for (int i = 0; i < HALO; i++) {
      for (int j = 0; j < smColDim; j++) {
        index = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
        shSrc[i * smColDim + j] = src[index];
        index = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
        shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[index];
      }
    }

    // Left and Right Halos
    for (int i = 0; i < HALO; i++) {
      for (int j = 0; j < smRowDim - HALO*2; j++) {
        index = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + blockIdx.y * blockDim.y * TILE_SIZE + i;
        shSrc[(HALO + j) * smColDim + i] = src[index];
        index = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
        shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[index];
      }
    }
  }

  // Synchronize to ensure all halo regions are loaded
  __syncthreads();

  // Stencil operation
  for (int i = 0; i < TILE_SIZE; i++) {
    for (int j = 0; j < TILE_SIZE; j++) {
      int globalIndex = (baseGlobalRow + i) * N + (baseGlobalCol + j);
      int shMemIndex = (HALO + threadIdx.x * TILE_SIZE + i) * smColDim + (HALO + threadIdx.y * TILE_SIZE + j);

      // Getting the neighbors
      double north = shSrc[shMemIndex - smColDim];
      double south = shSrc[shMemIndex + smColDim];
      double east = shSrc[shMemIndex + 1];
      double west = shSrc[shMemIndex - 1];

      // Real Stencil operation
      dst[globalIndex] = (north + south + east + west) / 5.5;
    }
  }
}