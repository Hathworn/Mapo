#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // SharedMem Columns and Rows Dimension
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Global indexes
    int globalIndex;
    int shMemIndex;

    // Calculate local (tile) start points in global memory
    int baseGlobalX = HALO * N + blockIdx.x * blockDim.x * TILE_SIZE * N + threadIdx.x * TILE_SIZE * N;
    int baseGlobalY = blockIdx.y * blockDim.y * TILE_SIZE + threadIdx.y * TILE_SIZE;

    // Copying inner part to shared memory
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            globalIndex = baseGlobalX + i * N + baseGlobalY + j + HALO;
            shMemIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + HALO + threadIdx.y * TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    __syncthreads();

    // Handle halos, execute only on the first thread of the block once
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;

        // Top and Bottom halos
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smColDim; j++) {
                indexTopHalo = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                indexBottomHalo = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }
        
        // Left and Right halos
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smRowDim - HALO * 2; j++) {
                indexLeftHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + blockIdx.y * blockDim.y * TILE_SIZE + i;
                indexRightHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Apply stencil
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            globalIndex = baseGlobalX + i * N + baseGlobalY + j + HALO;
            shMemIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + HALO + threadIdx.y * TILE_SIZE + j;

            // Fetch neighboring values
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];

            // Perform stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}