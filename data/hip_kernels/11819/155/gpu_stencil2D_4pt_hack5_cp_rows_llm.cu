#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt_hack5_cp_rows(double *dst, double *shared_cols, double *shared_rows, int tile_y, int M, int N) {
    // Calculate base indices for global memory access
    int base_global_row = tile_y * blockIdx.y;
    int base_global_col = blockDim.x * blockIdx.x;
    int base_global_idx = N * base_global_row + base_global_col;
    int nextRow = base_global_row + 1;
    int t = threadIdx.x;
    
    // Check boundary conditions
    bool legalCurCol = (base_global_col + t) < N;
    bool legalNextRow = (nextRow < M);
    
    // Determine shared memory indices
    int idx = (base_global_row / tile_y) * 2 * N + t + base_global_col;
    int idx_nextrow = idx + N;
    
    // Copy data to shared memory if within bounds
    if (legalCurCol) {
        shared_rows[idx] = dst[base_global_idx + t];
    }
    if (legalNextRow && legalCurCol) {
        shared_rows[idx_nextrow] = dst[base_global_idx + N + t];
    }
    __syncthreads();
}