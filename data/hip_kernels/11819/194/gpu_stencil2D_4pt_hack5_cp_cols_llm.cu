#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt_hack5_cp_cols(double * dst, double * shared_cols, double *shared_rows, int tile_x, int tile_y, int M, int N) {

#ifdef CUDA_DARTS_DEBUG
    if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.y == 0)) {
        printf("copy cols begin!\n");
    }
#endif

    // Calculate the base global indices for the current and the next column
    int base_global_row = tile_y * blockIdx.y;
    int base_global_col = tile_x * blockIdx.x;
    int base_global_idx = N * base_global_row + base_global_col;
    int nextCol = base_global_col + 1;

    // Cache legality of next column in a register
    bool legalNextCol = (nextCol < N);
    int t = threadIdx.y;
    
    // Efficient calculations using thread index
    int idx = 2 * M * blockIdx.x + t + base_global_row;
    int idx_nextCol = idx + M;
    bool legalCurRow = (base_global_row + t) < M;
    
    // Coalesced memory access for shared_cols
    if (legalCurRow) {
        shared_cols[idx] = dst[base_global_idx + t * N];
    }
    if (legalNextCol && legalCurRow) {
        shared_cols[idx_nextCol] = dst[base_global_idx + t * N + 1];
    }
    __syncthreads();

#ifdef CUDA_CUDA_DEBUG
    if (blockIdx.x == 1 && t < 5) {
        printf("addr: %d ,%f,\n", idx_nextCol, shared_cols[idx_nextCol]);
    }
#endif

#ifdef CUDA_DARTS_DEBUG
    if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.y == 0)) {
        printf("copy cols finish!\n");
    }
#endif
}