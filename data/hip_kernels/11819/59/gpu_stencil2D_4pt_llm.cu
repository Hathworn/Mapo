#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // SharedMem Dimensions
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Calculate the start indices for global and shared memory
    int baseGlobalIndex = HALO * N + blockIdx.x * blockDim.x * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE + HALO;
    int baseShMemIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + HALO + threadIdx.y * TILE_SIZE;

    // Optimize by calculating the loop bounds only once
    int tileSizeN = TILE_SIZE * N;
    int shMemStride = smColDim * TILE_SIZE;
    
    // Copying to shared memory
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = baseGlobalIndex + threadIdx.x * tileSizeN + i * N + threadIdx.y * TILE_SIZE + j;
            int shMemIndex = baseShMemIndex + i * smColDim + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Halos
    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        // For Bottom and top row
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smColDim; j++)
            {
                int indexTopHalo = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                int indexBottomHalo = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        // For right and left Columns
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smRowDim - HALO * 2; j++)
            {
                int indexLeftHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + i;
                int indexRightHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Stencil operation
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = baseGlobalIndex + threadIdx.x * tileSizeN + i * N + threadIdx.y * TILE_SIZE + j;
            int shMemIndex = baseShMemIndex + i * smColDim + j;

            // Getting the neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }

    __syncthreads();
}