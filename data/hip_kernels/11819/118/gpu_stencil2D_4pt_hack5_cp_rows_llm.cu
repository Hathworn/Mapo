#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt_hack5_cp_rows(double *dst, double *shared_cols, double *shared_rows, int tile_y, int M, int N) {

#ifdef CUDA_DARTS_DEBUG
    if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0)) {
        printf("copy rows begin!\n");
    }
#endif

    int base_global_row = (tile_y * blockIdx.y);
    int base_global_col = blockDim.x * blockIdx.x;
    int base_global_idx = N * base_global_row + base_global_col;
    int t = threadIdx.x;

    // Calculate indices for current and next row, check boundaries
    int idx = (base_global_row / tile_y) * 2 * N + t + base_global_col;
    int idx_nextrow = idx + N;
    int global_idx_t = base_global_idx + t;
    int global_idx_nextrow_t = base_global_idx + N + t;

    bool legalCurCol = (base_global_col + t) < N;
    bool legalNextRow = (base_global_row + 1) < M;

    // Coalesced memory access
    if (legalCurCol) {
        shared_rows[idx] = dst[global_idx_t];
    }
    if (legalNextRow && legalCurCol) {
        shared_rows[idx_nextrow] = dst[global_idx_nextrow_t];
    }
    __syncthreads();

#ifdef CUDA_CUDA_DEBUG
    if (blockIdx.y == 0 && blockIdx.x == 2 && (t == 0 || t == 1)) {
        printf("addr:%d, val = %f\n", idx_nextrow, shared_rows[idx_nextrow]);
    }
#endif

#ifdef CUDA_DARTS_DEBUG
    if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0)) {
        printf("copy rows finish!\n");
    }
#endif
}