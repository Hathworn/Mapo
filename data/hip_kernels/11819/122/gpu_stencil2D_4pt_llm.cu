#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // Calculate shared memory dimensions
    int smColDim = HALO*2 + blockDim.y*TILE_SIZE;
    int smRowDim = HALO*2 + blockDim.x*TILE_SIZE;

    // Calculate global and shared memory indices for threads
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int globalIndexOffset = HALO*N + bx*blockDim.x*TILE_SIZE*N + by*blockDim.y*TILE_SIZE + HALO;
    int shMemIndexOffset = HALO*smColDim + tx*smColDim*TILE_SIZE + HALO + ty*TILE_SIZE;

    // Copying to shared memory - Inner part
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = globalIndexOffset + tx*TILE_SIZE*N + i*N + j;
            int shMemIndex = shMemIndexOffset + i*smColDim + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Halos
    if (tx == 0 && ty == 0)
    {
        // For Bottom and top row
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smColDim; j++)
            {
                int indexTopHalo = (bx*blockDim.x*TILE_SIZE + i)*N + by*blockDim.y*TILE_SIZE + j;
                int indexBottomHalo = (HALO + (bx+1)*blockDim.x*TILE_SIZE)*N + by*blockDim.y*TILE_SIZE + j;
                shSrc[i*smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x*TILE_SIZE + i)*smColDim + j] = src[indexBottomHalo];
            }
        }

        // For right and left Columns
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smRowDim - HALO*2; j++)
            {
                int indexLeftHalo = (HALO + bx*blockDim.x*TILE_SIZE + j)*N + by*blockDim.y*TILE_SIZE + i;
                int indexRightHalo = (HALO + bx*blockDim.x*TILE_SIZE + j)*N + (by+1)*blockDim.y*TILE_SIZE + HALO + i;
                shSrc[(HALO + j)*smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1)*smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Processes the stencil calculation
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = globalIndexOffset + tx*TILE_SIZE*N + i*N + j;
            int shMemIndex = shMemIndexOffset + i*smColDim + j;

            // Getting the neighbors and calculating stencil operation
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];
            
            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }

    __syncthreads();
}