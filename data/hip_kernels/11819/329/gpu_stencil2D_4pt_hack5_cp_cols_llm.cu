#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt_hack5_cp_cols(double * dst, double * shared_cols, double * shared_rows, int tile_x, int tile_y, int M, int N) {

#ifdef CUDA_CUDA_DEBUG
    if((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.y == 0)) {
        printf("copy cols begin!\n");
    }
#endif

    int base_global_row = tile_y * blockIdx.y;
    int base_global_col = tile_x * blockIdx.x;
    int base_global_idx = N * base_global_row + base_global_col;
    int nextCol = base_global_col + 1;
    bool legalNextCol = (nextCol < N);
    int t = threadIdx.y;

    // Compute both indices at once to reduce code repetition
    int idx = 2 * M * blockIdx.x + t + base_global_row;
    int row_offset = base_global_idx + t * N;
    
    // Combine conditions to minimize branching and memory accesses
    if(base_global_row + t < M) {
        shared_cols[idx] = dst[row_offset]; // Load current column
        if (legalNextCol) {
            shared_cols[idx + M] = dst[row_offset + 1]; // Load next column
        }
    }

    __syncthreads();

#ifdef CUDA_CUDA_DEBUG
    if(blockIdx.x == 1 && t < 5) {
        printf("addr: %d ,%f,\n", idx_nextCol, shared_cols[idx_nextCol]);
    }
#endif

#ifdef CUDA_CUDA_DEBUG
    if((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.y == 0)) {
        printf("copy cols finish!\n");
    }
#endif
}