#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil37_hack2_cp_slices(double * dst, double * shared_rows, double *shared_cols,
                                              double *shared_slices,int d_xpitch,int d_ypitch,int d_zpitch,
                                              int s_xpitch,int s_ypitch, int s_zpitch, int n_rows, int n_cols,
                                              int n_slices, int tile_x,int tile_y, int tile_z){

#ifdef CUDA_DARTS_DEBUG
    if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
        printf("copy slices: begin!\n");
        printf("copy slices: n_cols=%d,n_rows=%d,n_slices=%d\n",n_cols,n_rows,n_slices);
        printf("copy slices: gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n",gridDim.x,gridDim.y,gridDim.z);
        printf("copy slices: blockDim.x=%d,blockDim.y=%d,blockDim.z=%d\n",blockDim.x,blockDim.y,blockDim.z);
        printf("copy slices: tile_x=%d,tile_y=%d,tile_z=%d\n",tile_x,tile_y,tile_z);
    }
#endif

    int base_global_slice = tile_z * blockIdx.z;
    int base_global_row   = tile_y * blockIdx.y;
    int base_global_col   = blockDim.x * blockIdx.x;

    // Combined calculations to reduce redundancy
    int d_area = d_ypitch*d_xpitch;
    int s_area = s_ypitch*s_xpitch;
    int base_global_idx = base_global_slice*d_area + base_global_row * d_xpitch + base_global_col;

    int nextSlice = base_global_slice + 1;
    bool legalNextSlice = (nextSlice < n_slices);
    int tx = threadIdx.x;
    bool legalCurCol = (base_global_col + tx) < n_cols;

    for(int ty = 0; ty < tile_y; ++ty){
        bool legalCurRow = (base_global_row + ty) < n_rows;
        // Calculate indices once, reuse them
        int s_idx = blockIdx.z * s_area * 2 + (base_global_row + ty) * s_xpitch + base_global_col + tx;
        int d_idx = base_global_idx + ty * d_xpitch + tx;
        
        if (legalCurCol && legalCurRow){
            shared_slices[s_idx] = dst[d_idx]; // Copy current slice
        }
        if (legalNextSlice && legalCurCol && legalCurRow){
            shared_slices[s_idx + s_area] = dst[d_idx + d_area]; // Copy next slice
        }
    }
    __syncthreads();

#ifdef CUDA_CUDA_DEBUG
    if(blockIdx.z == 0 && blockIdx.y == 0 && blockIdx.x == 0 ){
        if(threadIdx.x == 0 || threadIdx.x == 1 || threadIdx.x == 2){
            int addr  = s_xpitch * s_ypitch + blockDim.x * blockIdx.x + threadIdx.x;
            int addr1 = addr + s_xpitch;
            int addr2 = addr + s_xpitch * 2;

            int daddr  = d_xpitch * d_ypitch + blockDim.x * blockIdx.x + threadIdx.x;
            int daddr1 = daddr + d_xpitch;
            int daddr2 = daddr + d_xpitch * 2;
            printf("copy slices: blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,dst: addr= %d, val= %f\n", 
                    blockIdx.x, blockIdx.y, blockIdx.z, daddr, dst[daddr]);
            printf("copy slices: blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,dst: addr= %d, val= %f\n",
                    blockIdx.x, blockIdx.y, blockIdx.z, daddr1, dst[daddr1]);
            printf("copy slices: blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,dst: addr= %d, val= %f\n",
                    blockIdx.x, blockIdx.y, blockIdx.z, daddr2, dst[daddr2]);

            printf("copy slices: blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,shared_slices: addr= %d, val= %f\n",
                    blockIdx.x, blockIdx.y, blockIdx.z, addr, shared_slices[addr]);
            printf("copy slices: blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,shared_slices: addr= %d, val= %f\n",
                    blockIdx.x, blockIdx.y, blockIdx.z, addr1, shared_slices[addr1]);
            printf("copy slices: blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,shared_slices: addr= %d, val= %f\n",
                    blockIdx.x, blockIdx.y, blockIdx.z, addr2, shared_slices[addr2]);
        }
    }
#endif

#ifdef CUDA_DARTS_DEBUG
    if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
        printf("copy slices end!\n");
    }
#endif
}