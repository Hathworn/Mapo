#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // indexes
    int i, j;

    // neighbor's values
    double north, south, east, west;

    // SharedMem Columns Dimension
    int smColDim = HALO*2+blockDim.y*TILE_SIZE;
    int smRowDim = HALO*2+blockDim.x*TILE_SIZE;

    // Compute unique global index for each thread
    int globalBaseIndex = HALO*N + blockIdx.x * blockDim.x * TILE_SIZE * N + threadIdx.x * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE + threadIdx.y * TILE_SIZE + HALO;
    int shMemBaseIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + HALO + threadIdx.y * TILE_SIZE;

    // Copying to shared memory (Inner part)
    for (i = 0 ; i < TILE_SIZE ; i++) {
        for (j = 0 ; j < TILE_SIZE ; j++) {
            shSrc[shMemBaseIndex + i * smColDim + j] = src[globalBaseIndex + i * N + j];
        }
    }

    // Synchronize after loading inner part
    __syncthreads();

    // Handle halos with branch reduction
    if (threadIdx.x == 0 && threadIdx.y == 0) {

        int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;

        // Load top and bottom halo rows in shared memory
        for (i = 0 ; i < HALO ; i++) {
            for (j = 0 ; j < smColDim ; j++) {
                indexTopHalo = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                indexBottomHalo = (HALO + (blockIdx.x+1) * blockDim.x * TILE_SIZE) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        // Load left and right halo columns in shared memory
        for (i = 0 ; i < HALO ; i++) {
            for (j = 0 ; j < smRowDim - HALO * 2; j++) {
                indexLeftHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + i;
                indexRightHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    // Synchronize after loading halos
    __syncthreads();

    // Perform stencil computation
    for (i = 0 ; i < TILE_SIZE ; i++) {
        for (j = 0 ; j < TILE_SIZE ; j++) {
            int shMemIndex = shMemBaseIndex + i * smColDim + j;

            // Getting the neighbors
            north = shSrc[shMemIndex - smColDim];
            south = shSrc[shMemIndex + smColDim];
            east  = shSrc[shMemIndex + 1];
            west  = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalBaseIndex + i * N + j] = (north + south + east + west) / 5.5;
        }
    }

    // Synchronize before exiting the kernel
    __syncthreads();
}