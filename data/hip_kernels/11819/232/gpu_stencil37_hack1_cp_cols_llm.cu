#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil37_hack1_cp_cols(double *dst, double *shared_rows, double *shared_cols, double *shared_slices, int n_rows, int n_cols, int n_slices, int tile_x, int tile_y, int tile_z) {

#ifdef CUDA_DARTS_DEBUG
    if ((blockIdx.x == 0) && (blockIdx.y == 0) && (blockIdx.z == 0) && (threadIdx.y == 0) && (threadIdx.x == 0) && (threadIdx.z == 0)) {
        printf("copy cols: begin\n");
        printf("copy cols: gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n", gridDim.x, gridDim.y, gridDim.z);
        printf("copy cols: blockDim.x=%d,blockDim.y=%d,blockDim.z=%d\n", blockDim.x, blockDim.y, blockDim.z);
        printf("copy cols: tile_x=%d,tile_y=%d,tile_z=%d\n", tile_x, tile_y, tile_z);
    }
#endif

    int base_global_slice = tile_z * blockIdx.z;
    int base_global_row = blockDim.y * blockIdx.y;
    int base_global_col = tile_x * blockIdx.x;

    int area_dst = n_rows * n_cols;
    int area_shared = gridDim.x * n_rows * 2;

#ifdef CUDA_CUDA_DEBUG
    if ((blockIdx.x == 0) && (blockIdx.y == 0) && (blockIdx.z == 0) && (threadIdx.y == 0) && (threadIdx.x == 0) && (threadIdx.z == 0)) {
        printf("area_shared=%d\n", area_shared);
    }
#endif

    int base_global_idx = base_global_slice * area_dst + base_global_row * n_cols + base_global_col;
    int nextCol = base_global_col + 1;
    bool legalNextCol = (nextCol < n_cols);
    int ty = threadIdx.y;
    bool legalCurRow = (base_global_row + ty) < n_rows;

    int idx_base_shared = (base_global_slice) * area_shared + blockIdx.x * 2 * n_rows + blockIdx.y * blockDim.y;

    for (int tz = 0; tz < tile_z; ++tz) {
        bool legalCurSlice = (base_global_slice + tz) < n_slices;
        int idx_dst = base_global_idx + tz * area_dst + ty * n_cols;
        int idx_shared = idx_base_shared + ty + tz * area_shared;

        // Use condition branching to avoid warp divergence
        if (legalCurRow && legalCurSlice) {
            shared_cols[idx_shared] = dst[idx_dst];
            if (legalNextCol) {
                shared_cols[idx_shared + n_rows] = dst[idx_dst + 1];
            }
        }

        __syncthreads(); // Here is required if data in shared_cols is accessed by multiple threads
    }

    __syncthreads();

#ifdef CUDA_CUDA_DEBUG
    if (blockIdx.z == 0 && blockIdx.y == 0 && blockIdx.x == 0 && (threadIdx.x == 0)) {
        //printf("shared_cols: addr:%d, val = %f\n", threadIdx.y,shared_cols[threadIdx.y]);
    }
#endif

#ifdef CUDA_DARTS_DEBUG
    if ((blockIdx.x == 0) && (blockIdx.y == 0) && (blockIdx.z == 0) && (threadIdx.y == 0) && (threadIdx.x == 0) && (threadIdx.z == 0)) {
        printf("copy cols end!\n");
    }
#endif
}