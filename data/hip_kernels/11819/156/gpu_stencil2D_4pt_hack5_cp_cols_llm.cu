#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt_hack5_cp_cols(double * dst, double * shared_cols, double *shared_rows,int tile_x,int tile_y, int M, int N){

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.y==0)){
printf("copy cols begin!\n");
}
#endif

int base_global_row = tile_y  * blockIdx.y;
int base_global_col = tile_x  * blockIdx.x;
int base_global_idx = N*base_global_row + base_global_col ;
bool legalCurRow = (base_global_row + threadIdx.y)<M;
bool legalNextCol = (base_global_col + 1)<N;

int idx = 2*M*blockIdx.x + threadIdx.y + base_global_row;

// Optimize index calculation and memory access
if(legalCurRow) {
    shared_cols[idx] = dst[base_global_idx + threadIdx.y*N];
    if (legalNextCol) {
        shared_cols[idx + M] = dst[base_global_idx + threadIdx.y*N + 1];
    }
}
__syncthreads();

#ifdef CUDA_CUDA_DEBUG
if(blockIdx.x==1 && threadIdx.y<5){
printf("addr: %d, %f,\n", idx + M, shared_cols[idx + M]);
}
#endif

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.y==0)){
printf("copy cols finish!\n");
}
#endif
}