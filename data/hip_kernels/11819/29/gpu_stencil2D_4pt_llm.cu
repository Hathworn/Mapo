#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // SharedMem Columns Dimension
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Thread indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Calculate base global and shared memory indices
    int base_global_idx = HALO * N + blockIdx.x * blockDim.x * TILE_SIZE * N + tx * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE + ty * TILE_SIZE + HALO;
    int base_shared_idx = HALO * smColDim + tx * smColDim * TILE_SIZE + HALO + ty * TILE_SIZE;

    // Copying to shared memory - Inner part
    #pragma unroll
    for (int i = 0; i < TILE_SIZE; i++)
    {
        #pragma unroll
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = base_global_idx + i * N + j;
            int shMemIndex = base_shared_idx + i * smColDim + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Halos
    if (tx == 0 && ty == 0)
    {
        int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;
        // For Bottom and top row
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smColDim; j++)
            {
                indexTopHalo = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                indexBottomHalo = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        // For right and left Columns
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smRowDim - HALO * 2; j++)
            {
                indexLeftHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + i;
                indexRightHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Stencil operation
    #pragma unroll
    for (int i = 0; i < TILE_SIZE; i++)
    {
        #pragma unroll
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = base_global_idx + i * N + j;
            int shMemIndex = base_shared_idx + i * smColDim + j;

            // Get neighbor values
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}