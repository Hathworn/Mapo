#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // SharedMem Collumns Dimension
    int smColDim = HALO*2+blockDim.y*TILE_SIZE;
    int smRowDim = HALO*2+blockDim.x*TILE_SIZE;

    // Calculate thread's base indices
    int baseGlobalX = blockIdx.x * blockDim.x * TILE_SIZE;
    int baseGlobalY = blockIdx.y * blockDim.y * TILE_SIZE;

    // Linearize shared memory indices
    int threadBaseX = HALO + threadIdx.x * TILE_SIZE;
    int threadBaseY = HALO + threadIdx.y * TILE_SIZE;

    // Copy inner part to shared memory
    for (int i = 0; i < TILE_SIZE; ++i) {
        for (int j = 0; j < TILE_SIZE; ++j) {
            int globalIndex = (HALO + baseGlobalX + threadIdx.x * TILE_SIZE + i) * N 
                            + (HALO + baseGlobalY + threadIdx.y * TILE_SIZE + j);
            int shMemIndex = (threadBaseX + i) * smColDim + (threadBaseY + j);
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Synchronize to ensure all loads to shared memory are complete
    __syncthreads();

    // Copy halo regions only by the first thread
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;

        // Halo - Top and Bottom Rows
        for (int i = 0; i < HALO; ++i) {
            for (int j = 0; j < smColDim; ++j) {
                indexTopHalo = (baseGlobalX + i) * N + (baseGlobalY + j);
                indexBottomHalo = (HALO + baseGlobalX + blockDim.x * TILE_SIZE + i) * N + (baseGlobalY + j);
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        // Halo - Left and Right Columns
        for (int i = 0; i < HALO; ++i) {
            for (int j = 0; j < smRowDim - HALO*2; ++j) {
                indexLeftHalo = (HALO + baseGlobalX + j) * N + (baseGlobalY + i);
                indexRightHalo = (HALO + baseGlobalX + j) * N + (HALO + baseGlobalY + blockDim.y * TILE_SIZE + i);
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    // Synchronize to ensure halo region is loaded
    __syncthreads();

    // Stencil operation
    for (int i = 0; i < TILE_SIZE; ++i) {
        for (int j = 0; j < TILE_SIZE; ++j) {
            int globalIndex = (HALO + baseGlobalX + threadIdx.x * TILE_SIZE + i) * N 
                            + (HALO + baseGlobalY + threadIdx.y * TILE_SIZE + j);
            int shMemIndex = (threadBaseX + i) * smColDim + (threadBaseY + j);

            // Accessing neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}