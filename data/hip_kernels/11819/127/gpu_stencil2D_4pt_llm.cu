#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // Shared Memory Dimensions
    int smColDim = HALO*2+blockDim.y*TILE_SIZE;
    int smRowDim = HALO*2+blockDim.x*TILE_SIZE;

    // Global indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Copying to shared memory - Inner part
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = HALO*N + x*TILE_SIZE*N + i*N + y*TILE_SIZE + j + HALO;
            int shMemIndex = HALO*smColDim + threadIdx.x*smColDim*TILE_SIZE + i*smColDim + HALO + threadIdx.y*TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Halos - Load borders
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        int smSize = blockDim.x * TILE_SIZE + HALO;
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smColDim; j++) {
                int topIndex = (blockIdx.x*blockDim.x*TILE_SIZE + i)*N + (blockIdx.y*blockDim.y*TILE_SIZE) + j;
                int bottomIndex = (HALO + (blockIdx.x+1)*blockDim.x*TILE_SIZE)*N + (blockIdx.y*blockDim.y*TILE_SIZE) + j;
                shSrc[i*smColDim + j] = src[topIndex];
                shSrc[(smSize+i)*smColDim + j] = src[bottomIndex];
            }
        }
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smRowDim - HALO*2; j++) {
                int leftIndex = (HALO + blockIdx.x*blockDim.x*TILE_SIZE + j)*N + (blockIdx.y*blockDim.y*TILE_SIZE) + i;
                int rightIndex = (HALO + blockIdx.x*blockDim.x*TILE_SIZE + j)*N + ((blockIdx.y+1)*blockDim.y*TILE_SIZE) + HALO + i;
                shSrc[(HALO+j)*smColDim + i] = src[leftIndex];
                shSrc[(HALO+j+1)*smColDim - HALO + i] = src[rightIndex];
            }
        }
    }
    
    __syncthreads();

    // Compute stencil
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = HALO*N+x*TILE_SIZE*N+i*N+y*TILE_SIZE+j+HALO;
            int shMemIndex = HALO*smColDim+threadIdx.x*smColDim*TILE_SIZE+i*smColDim+HALO+threadIdx.y*TILE_SIZE+j;

            // Getting the neighbors
            double north = shSrc[shMemIndex-smColDim];
            double south = shSrc[shMemIndex+smColDim];
            double east  = shSrc[shMemIndex+1];
            double west  = shSrc[shMemIndex-1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
    
    __syncthreads();
}