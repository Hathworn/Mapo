#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // indexes
    int i, j;

    // neighbor's values
    double north, south, east, west;

    // SharedMem Columns Dimension
    int smColDim = HALO*2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO*2 + blockDim.x * TILE_SIZE;

    // Copying to shared memory
    // Inner part - optimized memory access pattern
    int baseGlobalIndex = HALO*N + blockIdx.x * blockDim.x * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE + HALO;
    int baseShMemIndex = HALO * smColDim + HALO;
    
    for (i = 0; i < TILE_SIZE; i++) {
        for (j = 0; j < TILE_SIZE; j++) {
            int globalIndex = baseGlobalIndex + threadIdx.x * TILE_SIZE * N + i * N + threadIdx.y * TILE_SIZE + j;
            int shMemIndex = baseShMemIndex + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + threadIdx.y * TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Halos - streamlined logic using optimized thread control
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;

        // For Bottom and top row
        for (i = 0; i < HALO; i++) {
            int baseHaloRowGlobalIndex = blockIdx.x * blockDim.x * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE;
            for (j = 0; j < smColDim; j++) {
                indexTopHalo = baseHaloRowGlobalIndex + i * N + j;
                indexBottomHalo = baseGlobalIndex + (HALO + blockDim.x * TILE_SIZE + i) * N + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        // For right and left Columns
        for (j = 0; j < smRowDim - HALO * 2; j++) {
            int baseHaloColGlobalIndex = HALO + blockIdx.x * blockDim.x * TILE_SIZE + j;
            indexLeftHalo = baseHaloColGlobalIndex * N + blockIdx.y * blockDim.y * TILE_SIZE;
            indexRightHalo = baseHaloColGlobalIndex * N + (blockIdx.y + 1) * blockDim.y * TILE_SIZE + HALO;
            shSrc[(HALO + j) * smColDim] = src[indexLeftHalo];
            shSrc[(HALO + j + 1) * smColDim - HALO] = src[indexRightHalo];
        }
    }

    __syncthreads();

    // Process stencil operation
    for (i = 0; i < TILE_SIZE; i++) {
        for (j = 0; j < TILE_SIZE; j++) {
            int globalIndex = baseGlobalIndex + threadIdx.x * TILE_SIZE * N + i * N + threadIdx.y * TILE_SIZE + j;
            int shMemIndex = baseShMemIndex + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + threadIdx.y * TILE_SIZE + j;

            // Getting the neighbors
            north = shSrc[shMemIndex - smColDim];
            south = shSrc[shMemIndex + smColDim];
            east = shSrc[shMemIndex + 1];
            west = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }

    __syncthreads();
}