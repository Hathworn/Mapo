#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double *dst, double *src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double *shSrc = shared_mem;

    // SharedMem Columns Dimension
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Calculate global indices
    int base_global_i = HALO * N + blockIdx.x * blockDim.x * TILE_SIZE * N;
    int base_global_j = HALO + blockIdx.y * blockDim.y * TILE_SIZE;
    int base_sh_i = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE;
    int base_sh_j = HALO + threadIdx.y * TILE_SIZE;

    // Calculate thread indices for the shared memory
    int thread_i = threadIdx.x * TILE_SIZE;
    int thread_j = threadIdx.y * TILE_SIZE;

    // Initialize shared memory with src values
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = base_global_i + thread_i * N + i * N + base_global_j + thread_j + j;
            int shMemIndex = base_sh_i + thread_i * smColDim + i * smColDim + base_sh_j + thread_j + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    __syncthreads(); // Sync threads after loading into shared memory

    // Load halos into shared memory
    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;
        // For Bottom and top row
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smColDim; j++)
            {
                indexTopHalo = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                indexBottomHalo = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        // For right and left Columns
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smRowDim - HALO * 2; j++)
            {
                indexLeftHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + i;
                indexRightHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads(); // Ensure all threads have completed halo loading

    // Perform stencil operations
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = base_global_i + thread_i * N + i * N + base_global_j + thread_j + j;
            int shMemIndex = base_sh_i + thread_i * smColDim + i * smColDim + base_sh_j + thread_j + j;

            // Getting the neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east = shSrc[shMemIndex + 1];
            double west = shSrc[shMemIndex - 1];
            
            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}