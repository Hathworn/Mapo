#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt_hack5_cp_cols(double * dst, double * shared_cols, double * shared_rows, int tile_x, int tile_y, int M, int N) {

#ifdef CUDA_DARTS_DEBUG
if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.y == 0)) {
    printf("copy cols begin!\n");
}
#endif

// Precompute indices to reduce redundant calculations
int base_global_idx = N * tile_y * blockIdx.y + tile_x * blockIdx.x;
int nextCol = base_global_idx + 1;
int shift = 2 * M * blockIdx.x + threadIdx.y;
int idx = shift + tile_y * blockIdx.y;
bool legalCurRow = (tile_y * blockIdx.y + threadIdx.y) < M;

if (legalCurRow) {
    shared_cols[idx] = dst[base_global_idx + threadIdx.y * N];
    if (nextCol < N) {
        shared_cols[idx + M] = dst[nextCol + threadIdx.y * N];
    }
}

// Ensure all threads are synchronized before proceeding
__syncthreads();

#ifdef CUDA_CUDA_DEBUG
if (blockIdx.x == 1 && threadIdx.y < 5) {
    printf("addr: %d, %f,\n", idx + M, shared_cols[idx + M]);
}
#endif

#ifdef CUDA_DARTS_DEBUG
if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.y == 0)) {
    printf("copy cols finish!\n");
}
#endif
}