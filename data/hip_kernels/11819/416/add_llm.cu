#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
    extern __shared__ int shared_mem[];
    int idx = threadIdx.x;

    // Efficiently copy data to shared memory
    shared_mem[idx] = idx;
    __syncthreads();

    // Compute addition using shared memory
    int val = shared_mem[idx];
    a[idx] = val;
    b[idx] = val;
    c[idx] = a[idx] + b[idx];
}