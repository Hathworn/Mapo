#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;
    
    // indexes
    int i, j;

    // neighbor's values
    double north, south, east, west;
    
    // SharedMem Columns Dimension
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;
    
    // Calculate global and shared memory indexes for each thread
    int globalBaseIndex = HALO * N + blockIdx.x * blockDim.x * TILE_SIZE * N + threadIdx.x * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE + threadIdx.y * TILE_SIZE + HALO;
    int sharedBaseIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + HALO + threadIdx.y * TILE_SIZE;

    // Copying to shared memory (Inner part)
    for (i = 0; i < TILE_SIZE; i++)
    {
        for (j = 0; j < TILE_SIZE; j++)
        {
            shSrc[sharedBaseIndex + i * smColDim + j] = src[globalBaseIndex + i * N + j];
        }
    }

    // Halos (Only the first thread handles the halos)
    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;

        // Top and bottom rows
        for (i = 0; i < HALO; i++)
        {
            for (j = 0; j < smColDim; j++)
            {
                indexTopHalo = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                indexBottomHalo = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE - HALO + i) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        // Left and right columns
        for (i = 0; i < HALO; i++)
        {
            for (j = HALO; j < smRowDim - HALO; j++)
            {
                indexLeftHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j - HALO) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + i;
                indexRightHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j - HALO) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE - HALO + i);
                shSrc[j * smColDim + i] = src[indexLeftHalo];
                shSrc[j * smColDim + (smColDim - HALO + i)] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Stencil computation
    for (i = 0; i < TILE_SIZE; i++)
    {
        for (j = 0; j < TILE_SIZE; j++)
        {
            int shMemIndex = sharedBaseIndex + i * smColDim + j;

            // Getting the neighbors
            north = shSrc[shMemIndex - smColDim];
            south = shSrc[shMemIndex + smColDim];
            east = shSrc[shMemIndex + 1];
            west = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalBaseIndex + i * N + j] = (north + south + east + west) / 5.5;
        }
    }
}