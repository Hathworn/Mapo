#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double *dst, double *src, int M, int N) {
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double *shSrc = shared_mem;

    // Calculate thread indices once
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;
    int t_offset = ty * TILE_SIZE + tx;

    // Shared memory dimensions
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    
    // Optimize global and shared memory indices calculations
    int globalIndexBase = HALO * N + bx * blockDim.x * TILE_SIZE * N + by * blockDim.y * TILE_SIZE;
    int shMemIndexBase = HALO * smColDim + tx * smColDim * TILE_SIZE + HALO;

    // Load data to shared memory
    for (int i = 0; i < TILE_SIZE; i++) {
        int gIndex = globalIndexBase + tx * TILE_SIZE * N + i * N + ty * TILE_SIZE + HALO;
        int sIndex = shMemIndexBase + i * smColDim + t_offset;
        shSrc[sIndex] = src[gIndex];
    }

    // Load halo regions using boundary threads
    if (tx == 0 && ty == 0) {
        int topHaloOffset = bx * blockDim.x * TILE_SIZE;
        int bottomHaloOffset = (HALO + (bx + 1) * blockDim.x * TILE_SIZE);

        for (int i = 0; i < HALO; ++i) {
            for (int j = 0; j < smColDim; ++j) {
                shSrc[i * smColDim + j] = src[(topHaloOffset + i) * N + by * blockDim.y * TILE_SIZE + j];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[(bottomHaloOffset + i) * N + by * blockDim.y * TILE_SIZE + j];
            }
        }

        int leftColOffset = HALO + bx * blockDim.x * TILE_SIZE;
        int rightColOffset = (HALO + (by + 1) * blockDim.y * TILE_SIZE + HALO);

        for (int i = 0; i < HALO; ++i) {
            for (int j = 0; j < smColDim - HALO * 2; ++j) {
                shSrc[(HALO + j) * smColDim + i] = src[(leftColOffset + j) * N + by * blockDim.y * TILE_SIZE + i];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[(leftColOffset + j) * N + rightColOffset + i];
            }
        }
    }

    __syncthreads();

    // Perform computation
    for (int i = 0; i < TILE_SIZE; i++) {
        int globalIndex = globalIndexBase + tx * TILE_SIZE * N + i * N + t_offset;
        int shMemIndex = shMemIndexBase + i * smColDim + t_offset;
        double north = shSrc[shMemIndex - smColDim];
        double south = shSrc[shMemIndex + smColDim];
        double east = shSrc[shMemIndex + 1];
        double west = shSrc[shMemIndex - 1];

        // Real stencil operation
        dst[globalIndex] = (north + south + east + west) / 5.5;
    }

    __syncthreads();
}