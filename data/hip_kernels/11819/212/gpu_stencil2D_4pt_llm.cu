#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double *dst, double *src, int M, int N) {
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double *shSrc = shared_mem;

    // Calculate global thread indices
    int global_i = blockIdx.x * blockDim.x + threadIdx.x;
    int global_j = blockIdx.y * blockDim.y + threadIdx.y;

    // Shared memory indices
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;
    int local_i = threadIdx.x;
    int local_j = threadIdx.y;

    int shMemIndex = (local_i + HALO) * smColDim + (local_j + HALO);

    // Load shared memory with halo data
    for (int i = -HALO; i < TILE_SIZE + HALO; i++) {
        for (int j = -HALO; j < TILE_SIZE + HALO; j++) {
            int i_global = global_i * TILE_SIZE + i;
            int j_global = global_j * TILE_SIZE + j;
            if (i_global >= 0 && i_global < M && j_global >= 0 && j_global < N) {
                shSrc[(local_i * TILE_SIZE + i + HALO) * smColDim + (local_j * TILE_SIZE + j + HALO)] =
                            src[i_global * N + j_global];
            }
        }
    }

    __syncthreads();

    // Perform stencil operation
    if (global_i < M / TILE_SIZE && global_j < N / TILE_SIZE) {
        for (int i = 0; i < TILE_SIZE; i++) {
            for (int j = 0; j < TILE_SIZE; j++) {
                double north = shSrc[shMemIndex - smColDim + i * smColDim + j];
                double south = shSrc[shMemIndex + smColDim + i * smColDim + j];
                double east  = shSrc[shMemIndex + 1 + i * smColDim + j];
                double west  = shSrc[shMemIndex - 1 + i * smColDim + j];
                dst[(global_i * TILE_SIZE + i) * N + (global_j * TILE_SIZE + j)] =
                    (north + south + east + west) / 5.5;
            }
        }
    }
}