#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // Compute indices for shared memory
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Calculate global and shared memory indices
    int baseGlobalIdx = HALO * N + blockIdx.x * blockDim.x * TILE_SIZE * N + threadIdx.x * TILE_SIZE * N + HALO;
    int baseShMemIdx = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + HALO;

    // Copy inner part to shared memory
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = baseGlobalIdx + i * N + blockIdx.y * blockDim.y * TILE_SIZE + threadIdx.y * TILE_SIZE + j;
            int shMemIndex = baseShMemIdx + i * smColDim + threadIdx.y * TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Copy halos into shared memory only by first thread of block
    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        int indexTop, indexBottom, indexLeft, indexRight;
        
        // For Bottom and top row halos
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smColDim; j++)
            {
                indexTop = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                indexBottom = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                shSrc[i * smColDim + j] = src[indexTop];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottom];
            }
        }
        
        // For right and left column halos
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smRowDim - HALO * 2; j++)
            {
                indexLeft = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + blockIdx.y * blockDim.y * TILE_SIZE + i;
                indexRight = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + (blockIdx.y + 1) * blockDim.y * TILE_SIZE + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeft];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRight];
            }
        }
    }

    __syncthreads();

    // Perform stencil operation
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = baseGlobalIdx + i * N + blockIdx.y * blockDim.y * TILE_SIZE + threadIdx.y * TILE_SIZE + j;
            int shMemIndex = baseShMemIdx + i * smColDim + threadIdx.y * TILE_SIZE + j;

            // Load neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];

            // Compute stencil result
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
    
    __syncthreads();
}