#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // SharedMem Dimensions
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Compute starting indices in global memory
    int base_index_x = blockIdx.x * blockDim.x * TILE_SIZE;
    int base_index_y = blockIdx.y * blockDim.y * TILE_SIZE;
    int thread_base_x = threadIdx.x * TILE_SIZE;
    int thread_base_y = threadIdx.y * TILE_SIZE;
    
    // Copying Inner part to shared memory
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = HALO * N + base_index_x * N + thread_base_x * N + i * N + base_index_y + thread_base_y + j + HALO;
            int shMemIndex = HALO * smColDim + thread_base_x * smColDim + i * smColDim + HALO + thread_base_y + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Synchronize to ensure all threads have copied their elements
    __syncthreads();

    // Thread 0 handles halo region loading
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        // Load top and bottom halos
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smColDim; j++) {
                int indexTopHalo = base_index_x * N + i * N + base_index_y + j;
                int indexBottomHalo = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + base_index_y + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }
        // Load left and right halos
        for (int j = 0; j < smRowDim - HALO * 2; j++) {
            for (int i = 0; i < HALO; i++) {
                int indexLeftHalo = (HALO + base_index_x + j) * N + base_index_y + i;
                int indexRightHalo = (HALO + base_index_x + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    // Synchronize to ensure halo regions are loaded
    __syncthreads();

    // Compute stencil
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = HALO * N + base_index_x * N + thread_base_x * N + i * N + base_index_y + thread_base_y + j + HALO;
            int shMemIndex = HALO * smColDim + thread_base_x * smColDim + i * smColDim + HALO + thread_base_y + j;

            // Getting the neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }

    // Synchronize before finishing kernel to ensure all computations are done
    __syncthreads();
}