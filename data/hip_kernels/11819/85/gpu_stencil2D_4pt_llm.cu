#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double* dst, double* src, int M, int N) {
    extern __shared__ double shared_mem[];
    double* shSrc = shared_mem;

    // Shared memory dimensions
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Calculate global and shared memory indices
    int global_i = blockIdx.x * blockDim.x * TILE_SIZE + threadIdx.x * TILE_SIZE + HALO;
    int global_j = blockIdx.y * blockDim.y * TILE_SIZE + threadIdx.y * TILE_SIZE + HALO;
    int shMem_i = HALO + threadIdx.x * TILE_SIZE;
    int shMem_j = HALO + threadIdx.y * TILE_SIZE;

    // Load inner part to shared memory
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = (global_i + i) * N + global_j + j;
            int shMemIndex = (shMem_i + i) * smColDim + shMem_j + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Load halos into shared memory
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        // Top and bottom halos
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smColDim; j++) {
                int topIndex = (global_i - HALO + i) * N + global_j - HALO + j;
                int bottomIndex = (global_i + blockDim.x * TILE_SIZE + i) * N + global_j - HALO + j;
                shSrc[i * smColDim + j] = src[topIndex];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[bottomIndex];
            }
        }

        // Left and right halos
        for (int i = 0; i < smRowDim - HALO * 2; i++) {
            for (int j = 0; j < HALO; j++) {
                int leftIndex = (global_i - HALO + i) * N + global_j - HALO + j;
                int rightIndex = (global_i - HALO + i) * N + global_j + blockDim.y * TILE_SIZE + j;
                shSrc[(HALO + i) * smColDim + j] = src[leftIndex];
                shSrc[(HALO + i + 1) * smColDim - HALO + j] = src[rightIndex];
            }
        }
    }
    
    __syncthreads();

    // Compute result
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = (global_i + i) * N + global_j + j;
            int shMemIndex = (shMem_i + i) * smColDim + shMem_j + j;

            // Fetch neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east = shSrc[shMemIndex + 1];
            double west = shSrc[shMemIndex - 1];

            // Stencil computation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}