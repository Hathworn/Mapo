#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double *dst, double *src, int M, int N) {
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double *shSrc = shared_mem;

    // Calculate indices and shared memory dimensions
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;
    int local_i = threadIdx.x * TILE_SIZE;
    int local_j = threadIdx.y * TILE_SIZE;
    
    // Calculate global and shared memory indices
    int baseGlobalIndex = HALO * N + blockIdx.x * blockDim.x * TILE_SIZE * N + 
                          blockIdx.y * blockDim.y * TILE_SIZE + HALO;
    int baseShMemIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + HALO;

    // Copy inner part to shared memory
    for (int i = 0; i < TILE_SIZE; ++i) {
        for (int j = 0; j < TILE_SIZE; ++j) {
            int globalIndex = baseGlobalIndex + local_i * N + local_j + i * N + j;
            int shMemIndex = baseShMemIndex + local_i * smColDim + local_j + i * smColDim + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }
    
    // Synchronize after loading inner region
    __syncthreads();

    // Load halo regions
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        for (int i = 0; i < HALO; ++i) {
            for (int j = 0; j < smColDim; ++j) {
                int indexTopHalo = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                int indexBottomHalo = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        for (int i = 0; i < HALO; ++i) {
            for (int j = 0; j < smRowDim - HALO * 2; ++j) {
                int indexLeftHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + i;
                int indexRightHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }
    
    // Synchronize after loading halo regions
    __syncthreads();

    // Compute stencil
    for (int i = 0; i < TILE_SIZE; ++i) {
        for (int j = 0; j < TILE_SIZE; ++j) {
            int globalIndex = baseGlobalIndex + local_i * N + local_j + i * N + j;
            int shMemIndex = baseShMemIndex + local_i * smColDim + local_j + i * smColDim + j;
            
            // Get the neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];

            // Real stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
    
    // Final synchronization
    __syncthreads();
}