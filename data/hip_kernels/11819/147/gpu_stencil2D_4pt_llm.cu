#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double *dst, double *src, int M, int N) {
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double *shSrc = shared_mem;

    // SharedMem Columns and Rows Dimension
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Initialize thread shared memory copy positions
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Global and shared memory indices
    int globalIndex, shMemIndex;

    // Copying to shared memory: Inner part
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            globalIndex = HALO * N + bx * blockDim.x * TILE_SIZE * N + tx * TILE_SIZE * N + i * N + by * blockDim.y * TILE_SIZE + ty * TILE_SIZE + j + HALO;
            shMemIndex = HALO * smColDim + tx * smColDim * TILE_SIZE + i * smColDim + HALO + ty * TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    __syncthreads();

    // Halos: Copying Top, Bottom, Left and Right halos by a single thread for efficiency
    if (tx == 0 && ty == 0) {
        int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;

        // Top and Bottom rows
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smColDim; j++) {
                indexTopHalo = (bx * blockDim.x * TILE_SIZE + i) * N + (by * blockDim.y * TILE_SIZE) + j;
                indexBottomHalo = (HALO + (bx + 1) * blockDim.x * TILE_SIZE) * N + (by * blockDim.y * TILE_SIZE) + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        // Left and Right columns
        for (int j = 0; j < smRowDim - HALO * 2; j++) {
            for (int i = 0; i < HALO; i++) {
                indexLeftHalo = (HALO + bx * blockDim.x * TILE_SIZE + j) * N + (by * blockDim.y * TILE_SIZE) + i;
                indexRightHalo = (HALO + bx * blockDim.x * TILE_SIZE + j) * N + ((by + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Stencil operation
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            globalIndex = HALO * N + bx * blockDim.x * TILE_SIZE * N + tx * TILE_SIZE * N + i * N + by * blockDim.y * TILE_SIZE + ty * TILE_SIZE + j + HALO;
            shMemIndex = HALO * smColDim + tx * smColDim * TILE_SIZE + i * smColDim + HALO + ty * TILE_SIZE + j;

            // Getting the neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east = shSrc[shMemIndex + 1];
            double west = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}