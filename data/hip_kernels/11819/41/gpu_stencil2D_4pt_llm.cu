#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // Compute thread's global index
    int global_i = blockIdx.x * blockDim.x * TILE_SIZE + threadIdx.x * TILE_SIZE;
    int global_j = blockIdx.y * blockDim.y * TILE_SIZE + threadIdx.y * TILE_SIZE;

    // SharedMem Columns and Rows Dimension
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Iterate over tile size block
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = global_i * N + i * N + global_j + j + HALO;
            int shMemIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + HALO + threadIdx.y * TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Load haloes into shared memory
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        int indexHalo, shMemIndex;
        // Top and Bottom Halos
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smColDim; j++) {
                indexHalo = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                shSrc[i * smColDim + j] = src[indexHalo];
                indexHalo = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexHalo];
            }
        }
        // Left and Right Halos
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smRowDim - HALO * 2; j++) {
                indexHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + blockIdx.y * blockDim.y * TILE_SIZE + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexHalo];
                indexHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexHalo];
            }
        }
    }

    __syncthreads();

    // Perform stencil computation
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = global_i * N + i * N + global_j + j + HALO;
            int shMemIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + HALO + threadIdx.y * TILE_SIZE + j;
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east = shSrc[shMemIndex + 1];
            double west = shSrc[shMemIndex - 1];
            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 4.0;
        }
    }

    __syncthreads();
}