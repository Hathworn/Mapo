#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double *dst, double *src, int M, int N)
{
    // Declare shared memory with dynamic size
    extern __shared__ double shared_mem[];
    double *shSrc = shared_mem;

    // Calculate shared memory dimensions once
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Calculate global indices for this block
    int globalIndexOffset = HALO * N + blockIdx.x * blockDim.x * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE;

    // Shared memory loading, optimized loop boundaries
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = globalIndexOffset + threadIdx.x * TILE_SIZE * N + i * N + threadIdx.y * TILE_SIZE + j + HALO;
            int shMemIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + HALO + threadIdx.y * TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Load halos if first thread
    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smColDim; j++)
            {
                // Calculate halo indices once
                int indexTopHalo = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                int indexBottomHalo = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                // Load top and bottom rows directly into shared memory
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }

            for (int j = 0; j < smRowDim - HALO * 2; j++)
            {
                int indexLeftHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + blockIdx.y * blockDim.y * TILE_SIZE + i;
                int indexRightHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                // Load left and right columns into shared memory
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Compute stencil, optimized loop boundaries
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = globalIndexOffset + threadIdx.x * TILE_SIZE * N + i * N + threadIdx.y * TILE_SIZE + j + HALO;
            int shMemIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + HALO + threadIdx.y * TILE_SIZE + j;

            // Get neighbor values
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east = shSrc[shMemIndex + 1];
            double west = shSrc[shMemIndex - 1];

            // Perform stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}