#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float * __restrict__ A, const float * __restrict__ B, float * __restrict__ C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = __ldg(&A[i]) + __ldg(&B[i]);  // Use __ldg for read-only data
    }
}