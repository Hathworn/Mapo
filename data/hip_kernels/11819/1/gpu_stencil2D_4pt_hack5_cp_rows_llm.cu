#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
#define ROTATE_DOWN(val,MAX) ((val-1==-1)?MAX-1:val-1)
#define ROTATE_UP(val,MAX) ((val+1)%MAX)
/**
* GPU Device kernel for the for 2D stencil
* First attempt during hackaton
* M = Rows, N = Cols INCLUDING HALOS
* In this version now we replace the size of the shared memory to be just 3 rows (actually 1+HALO*2) rows
*/

__global__ void gpu_stencil2D_4pt_hack5_cp_rows(double * dst, double * shared_cols, double *shared_rows,int tile_y,int M, int N){

#ifdef CUDA_DARTS_DEBUG
    if((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0)){
        printf("copy rows begin!\n");
    }
#endif
    
    int base_global_row = tile_y * blockIdx.y;
    int base_global_col = blockDim.x * blockIdx.x;
    int base_global_idx = N * base_global_row + base_global_col;
    int nextRow = base_global_row + 1;
    bool legalNextRow = (nextRow < M);
    int t = threadIdx.x;
    bool legalCurCol = (base_global_col + t) < N;
    int idx = (base_global_row / tile_y) * 2 * N + t + base_global_col;
    int idx_nextrow = idx + N;
    
    // Use registers to reduce global memory access
    double val_curRow = (legalCurCol) ? dst[base_global_idx + t] : 0.0; 
    double val_nextRow = (legalNextRow && legalCurCol) ? dst[base_global_idx + N + t] : 0.0;

    if (legalCurCol) {
        shared_rows[idx] = val_curRow;
    }
    if (legalNextRow && legalCurCol) {
        shared_rows[idx_nextrow] = val_nextRow;
    }
    __syncthreads();

#ifdef CUDA_CUDA_DEBUG
    if(blockIdx.y == 0 && blockIdx.x == 2 && (t == 0 || t == 1)){
        printf("addr:%d, val = %f\n", idx_nextrow, shared_rows[idx_nextrow]);
    }
#endif

#ifdef CUDA_DARTS_DEBUG
    if((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0)){
        printf("copy rows finish!\n");
    }
#endif
}