#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt_hack5_cp_cols(double * dst, double * shared_cols, double *shared_rows,int tile_x,int tile_y, int M, int N){

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.y==0)){
printf("copy cols begin!\n");
}
#endif

int base_global_row = tile_y * blockIdx.y;
int base_global_col = tile_x * blockIdx.x;
int base_global_idx = N * base_global_row + base_global_col;
int t = threadIdx.y;
int idx = 2 * M * blockIdx.x + t + base_global_row;
bool legalCurRow = (base_global_row + t) < M;

// Efficiently calculate address offsets
int currentColOffset = base_global_idx + t * N;
int nextColOffset = currentColOffset + 1;
bool legalNextCol = (base_global_col + 1) < N;

if (legalCurRow) {
    shared_cols[idx] = dst[currentColOffset];
}
if (legalNextCol && legalCurRow) {
    shared_cols[idx + M] = dst[nextColOffset];
}
__syncthreads();

#ifdef CUDA_CUDA_DEBUG
if(blockIdx.x == 1 && t < 5){
printf("addr: %d ,%f,\n",idx + M, shared_cols[idx + M]);
}
#endif

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.y == 0)){
printf("copy cols finish!\n");
}
#endif
}