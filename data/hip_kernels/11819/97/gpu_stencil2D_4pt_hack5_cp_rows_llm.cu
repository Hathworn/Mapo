#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt_hack5_cp_rows(double * dst, double * shared_cols, double *shared_rows, int tile_y, int M, int N){
    
    int base_global_row = tile_y * blockIdx.y;
    int base_global_col = blockDim.x * blockIdx.x;
    int base_global_idx = N * base_global_row + base_global_col;
    int nextRow = base_global_row + 1;
    int t = threadIdx.x;
    bool legalCurCol = (base_global_col + t) < N;
    
    // Calculate index for copying current and next row
    int idx = (base_global_row/tile_y) * 2 * N + t + base_global_col;
    int idx_nextrow = idx + N;
    
    // Copy current row elements if within bounds
    if (legalCurCol) {
        shared_rows[idx] = dst[base_global_idx + t];
    }
    
    // Copy next row elements if within legal range
    if (nextRow < M && legalCurCol) {
        shared_rows[idx_nextrow] = dst[base_global_idx + N + t];
    }

    __syncthreads();
}