#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N) {
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // indexes
    int i, j;

    // neighbor's values
    double north, south, east, west;

    // SharedMem Columns Dimension
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Local thread indices
    int local_i = threadIdx.x * TILE_SIZE;
    int local_j = threadIdx.y * TILE_SIZE;

    // Global indices
    int base_i = blockIdx.x * blockDim.x * TILE_SIZE + local_i + HALO;
    int base_j = blockIdx.y * blockDim.y * TILE_SIZE + local_j + HALO;

    // Copying to shared memory - Inner part
    for (i = 0; i < TILE_SIZE; i++) {
        for (j = 0; j < TILE_SIZE; j++) {
            int globalIndex = (base_i + i) * N + (base_j + j);
            int shMemIndex = (local_i + i + HALO) * smColDim + local_j + j + HALO;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    __syncthreads();

    // Only the first thread in the block handles the halo
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        // Handling Halos
        int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;

        // For Bottom and top row
        for (i = 0; i < HALO; i++) {
            for (j = 0; j < smColDim; j++) {
                indexTopHalo = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                indexBottomHalo = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        // For right and left Columns
        for (i = 0; i < HALO; i++) {
            for (j = 0; j < smRowDim - HALO * 2; j++) {
                indexLeftHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + blockIdx.y * blockDim.y * TILE_SIZE + i;
                indexRightHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Compute the stencil operation
    for (i = 0; i < TILE_SIZE; i++) {
        for (j = 0; j < TILE_SIZE; j++) {
            int globalIndex = (base_i + i) * N + (base_j + j);
            int shMemIndex = (local_i + i + HALO) * smColDim + local_j + j + HALO;

            // Getting the neighbors
            north = shSrc[shMemIndex - smColDim];
            south = shSrc[shMemIndex + smColDim];
            east  = shSrc[shMemIndex + 1];
            west  = shSrc[shMemIndex - 1];
            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}