#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
    extern __shared__ int shared_mem[];
    int * shmem = shared_mem;
    
    int tid = threadIdx.x;  // Use a temporary variable for thread index

    shmem[tid] = tid;  // Simplified to reuse the computed value
    int value = shmem[tid];  // Store shared memory value in a local variable

    a[tid] = value;  // Use local variable instead of multiple shared memory accesses
    b[tid] = value;  // Use local variable instead of multiple shared memory accesses
    c[tid] = a[tid] + b[tid];  // Sum values in registers
}