#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // Global thread coordinates
    int baseX = blockIdx.x * blockDim.x * TILE_SIZE + threadIdx.x * TILE_SIZE;
    int baseY = blockIdx.y * blockDim.y * TILE_SIZE + threadIdx.y * TILE_SIZE;

    // SharedMem extent
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Linearize indices
    int linearBlock = blockIdx.x * blockDim.x * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE;
    int linearThread = threadIdx.x * TILE_SIZE * N + threadIdx.y * TILE_SIZE;

    // Load inner elements into shared memory
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = HALO * N + linearBlock + linearThread + i * N + j + HALO;
            int shMemIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + HALO + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Load halos (top, bottom, left, right) into shared memory
    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        // Top and bottom rows
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smColDim; j++)
            {
                int indexTopHalo = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                int indexBottomHalo = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        // Left and right columns
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smRowDim - HALO * 2; j++)
            {
                int indexLeftHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + blockIdx.y * blockDim.y * TILE_SIZE + i;
                int indexRightHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Perform stencil computation
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = HALO * N + linearBlock + linearThread + i * N + j + HALO;
            int shMemIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + HALO + j;

            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }

    __syncthreads();
}