#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
#define ROTATE_DOWN(val,MAX) ((val-1==-1)?MAX-1:val-1)
#define ROTATE_UP(val,MAX) ((val+1)%MAX)

__global__ void gpu_stencil2D_4pt_hack5_cp_cols(double * dst, double * shared_cols, double *shared_rows, int tile_x, int tile_y, int M, int N) {

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.y==0)){
printf("copy cols begin!\n");
}
#endif

int base_global_row = tile_y * blockIdx.y;
int base_global_col = tile_x * blockIdx.x;
int base_global_idx = N * base_global_row + base_global_col;
int t = threadIdx.y;
int idx = 2 * M * blockIdx.x + t + base_global_row;
bool legalCurRow = (base_global_row + t) < M;

// Load column data to shared memory
if (legalCurRow) {
    shared_cols[idx] = dst[base_global_idx + t * N];
    if ((base_global_col + 1) < N) { // Combine condition checks into a single if
        shared_cols[idx + M] = dst[base_global_idx + t * N + 1];
    }
}

__syncthreads();

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.y==0)){
printf("copy cols finish!\n");
}
#endif
}