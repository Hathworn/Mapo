#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // indexes
    int i, j;

    // neighbor's values
    double north, south, east, west;

    // SharedMem Columns Dimension
    int smColDim = HALO*2+blockDim.y*TILE_SIZE;
    int smRowDim = HALO*2+blockDim.x*TILE_SIZE;

    // Copying to shared memory

    // Inner part
    // Use one-dimensional thread index and grid index calculations to improve performance
    int globalOffset = HALO*N + blockIdx.x * blockDim.x * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE;
    int shMemOffset = HALO * smColDim + HALO;

    for (i = 0; i < TILE_SIZE; i++) {
        for (j = 0; j < TILE_SIZE; j++) {
            int globalIndex = globalOffset + threadIdx.x * TILE_SIZE * N + i * N + threadIdx.y * TILE_SIZE + j;
            int shMemIndex = shMemOffset + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + threadIdx.y * TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    __syncthreads();

    // Handles Halo regions
    if (threadIdx.x == 0 && threadIdx.y == 0) {

        // Use one-dimensional loop index for optimization
        int topHaloOffset = blockIdx.x * blockDim.x * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE;
        int bottomHaloOffset = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + blockIdx.y * blockDim.y * TILE_SIZE;
        int leftHaloOffset = HALO + blockIdx.x * blockDim.x * TILE_SIZE;
        int rightHaloOffset = HALO + blockIdx.x * blockDim.x * TILE_SIZE + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE)+HALO;

        // For bottom and top row
        for (i = 0; i < HALO; i++) {
            for (j = 0; j < smColDim; j++) {
                shSrc[i * smColDim + j] = src[topHaloOffset + i * N + j];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[bottomHaloOffset + i * N + j];
            }
        }

        // For right and left Columns
        for (i = 0; i < HALO; i++) {
            for (j = 0; j < smRowDim - HALO * 2; j++) {
                shSrc[(HALO + j) * smColDim + i] = src[(leftHaloOffset + j) * N + blockIdx.y * blockDim.y * TILE_SIZE + i];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[(rightHaloOffset + j) * N + blockIdx.y * blockDim.y * TILE_SIZE + HALO + i];
            }
        }
    }

    __syncthreads();

    // Perform stencil computation
    for (i = 0; i < TILE_SIZE; i++) {
        for (j = 0; j < TILE_SIZE; j++) {
            int globalIndex = globalOffset + threadIdx.x * TILE_SIZE * N + i * N + threadIdx.y * TILE_SIZE + j;
            int shMemIndex = shMemOffset + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + threadIdx.y * TILE_SIZE + j;

            // Getting the neighbors
            north = shSrc[shMemIndex - smColDim];
            south = shSrc[shMemIndex + smColDim];
            east  = shSrc[shMemIndex + 1];
            west  = shSrc[shMemIndex - 1];
            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
            // dst[globalIndex] = (north + south + east + west) / 4;
        }
    }

    __syncthreads();
}
```
