#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt_hack5_cp_cols(double * dst, double * shared_cols, double *shared_rows, int tile_x, int tile_y, int M, int N) {

#ifdef CUDA_DARTS_DEBUG
if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.y == 0)) {
    printf("copy cols begin!\n");
}
#endif

int base_global_row = tile_y * blockIdx.y;
int base_global_col = tile_x * blockIdx.x;
int base_global_idx = N * base_global_row + base_global_col;
int t = threadIdx.y;
int idx = 2 * M * blockIdx.x + t + base_global_row;

// Using register to save repeated calculation
int dst_base_row = base_global_idx + t * N;

if (base_global_row + t < M) {
    shared_cols[idx] = dst[dst_base_row];
    if (base_global_col + 1 < N) {  // Combine conditions to reduce branching
        shared_cols[idx + M] = dst[dst_base_row + 1];
    }
}
__syncthreads();

#ifdef CUDA_CUDA_DEBUG
if (blockIdx.x == 1 && t < 5) {
    printf("addr: %d ,%f,\n", idx + M, shared_cols[idx + M]);
}
#endif

#ifdef CUDA_DARTS_DEBUG
if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.y == 0)) {
    printf("copy cols finish!\n");
}
#endif
}