#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
#define ROTATE_DOWN(val,MAX) ((val-1==-1)?MAX-1:val-1)
#define ROTATE_UP(val,MAX) ((val+1)%MAX)

// Optimize by using memory coalescing and reducing __syncthreads() calls
__global__ void gpu_stencil2D_4pt_hack2(double * dst, double * src, int M, int N)
{
    __shared__ double shared_mem[1 + HALO*2][GRID_TILE_X + HALO*2];

    int base_global_idx = N * (GRID_TILE_Y * blockIdx.y) + GRID_TILE_X * blockIdx.x;
    int thread_id = threadIdx.x;
    int center = 1, north = 0, south = 2;

    // Load initial shared memory
    for (int i = 0; i < 1 + HALO*2; i++) {
        for (int j = thread_id; j < GRID_TILE_X + 2*HALO; j += blockDim.x) {
            shared_mem[i][j] = src[base_global_idx + i*N + j];
        }
    }
    __syncthreads();

    // Main computation loop
    for (int curRow = HALO; curRow < GRID_TILE_Y; curRow++) {
        for (int j = thread_id + HALO; j < GRID_TILE_X + HALO; j += blockDim.x) {
            dst[base_global_idx + curRow*N + j] = (
                shared_mem[north][j] +
                shared_mem[south][j] +
                shared_mem[center][j-1] +
                shared_mem[center][j+1]
            ) / 5.5;
        }

        if (curRow + 2 < GRID_TILE_Y + HALO) {
            for (int j = thread_id; j < GRID_TILE_X + 2*HALO; j += blockDim.x) {
                shared_mem[north][j] = src[base_global_idx + (curRow + 2)*N + j];
            }
        }
        __syncthreads();

        // Rotate indices
        center = ROTATE_UP(center, 3);
        south = ROTATE_UP(south, 3);
        north = ROTATE_UP(north, 3);
    }

    // Final computation on the last row
    for (int j = thread_id + HALO; j < GRID_TILE_X + HALO; j += blockDim.x) {
        dst[base_global_idx + GRID_TILE_Y*N + j] = (
            shared_mem[north][j] +
            shared_mem[south][j] +
            shared_mem[center][j-1] +
            shared_mem[center][j+1]
        ) / 5.5;
    }
}