#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double *dst, double *src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double *shSrc = shared_mem;

    // indexes
    int i, j;

    // neighbor's values
    double north, south, east, west;

    // SharedMem Columns Dimension
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Calculate global and shared memory indices at once
    int baseGlobalIndex = HALO * N + blockIdx.x * blockDim.x * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE + HALO;
    int baseShMemIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + HALO;

    // Copying to shared memory
    // Inner part: use individual threads to copy their region
    for (i = 0; i < TILE_SIZE; ++i)
    {
        for (j = 0; j < TILE_SIZE; ++j)
        {
            int globalIndex = baseGlobalIndex + threadIdx.x * TILE_SIZE * N + i * N + threadIdx.y * TILE_SIZE + j;
            int shMemIndex = baseShMemIndex + i * smColDim + threadIdx.y * TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Synchronize before copying halos
    __syncthreads();

    // Halos
    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;

        // Using direct blocks to fill halos
        int blockOffset = blockIdx.x * blockDim.x * TILE_SIZE;
        int nextBlockOffset = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N;

        // For Bottom and top rows
        for (i = 0; i < HALO; ++i)
        {
            for (j = 0; j < smColDim; ++j)
            {
                indexTopHalo = blockOffset * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j + i * N;
                indexBottomHalo = nextBlockOffset + j + i * N;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        // For right and left Columns
        for (i = 0; i < HALO; ++i)
        {
            for (j = 0; j < smRowDim - HALO * 2; ++j)
            {
                int rowOffset = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + (blockIdx.y * blockDim.y * TILE_SIZE);
                indexLeftHalo = rowOffset + i;
                indexRightHalo = rowOffset + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    // Synchronize to ensure all halo data is up-to-date
    __syncthreads();

    // Perform stencil operation
    for (i = 0; i < TILE_SIZE; ++i)
    {
        for (j = 0; j < TILE_SIZE; ++j)
        {
            int globalIndex = baseGlobalIndex + threadIdx.x * TILE_SIZE * N + i * N + threadIdx.y * TILE_SIZE + j;
            int shMemIndex = baseShMemIndex + i * smColDim + threadIdx.y * TILE_SIZE + j;

            // Getting the neighbors
            north = shSrc[shMemIndex - smColDim];
            south = shSrc[shMemIndex + smColDim];
            east  = shSrc[shMemIndex + 1];
            west  = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 4.0; // Adjusted average denominator
        }
    }
}
```
