#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt_hack5_cp_rows(double * dst, double * shared_cols, double *shared_rows, int tile_y, int M, int N) {

#ifdef CUDA_DARTS_DEBUG
    if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0)) {
        printf("copy rows begin!\n");
    }
#endif

    int base_global_row = tile_y * blockIdx.y;
    int base_global_col = blockDim.x * blockIdx.x;
    int base_global_idx = N * base_global_row + base_global_col;
    int nextRow = base_global_row + 1;
    int t = threadIdx.x;

    // Store common calculations
    int global_col_within_bounds = base_global_col + t;

    // Check for valid column and row within bounds
    bool legalCurCol = global_col_within_bounds < N;
    bool legalNextRow = nextRow < M;

    int idx_base = (base_global_row / tile_y) * 2 * N + global_col_within_bounds;
    int idx_nextrow = idx_base + N;

    if (legalCurCol) {
        shared_rows[idx_base] = dst[base_global_idx + t];
    }
    if (legalNextRow && legalCurCol) {
        shared_rows[idx_nextrow] = dst[base_global_idx + N + t];
    }
    
    __syncthreads();

#ifdef CUDA_CUDA_DEBUG
    if (blockIdx.y == 0 && blockIdx.x == 2 && (t == 0 || t == 1)) {
        printf("addr:%d, val = %f\n", idx_nextrow, shared_rows[idx_nextrow]);
    }
#endif

#ifdef CUDA_DARTS_DEBUG
    if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0)) {
        printf("copy rows finish!\n");
    }
#endif
}