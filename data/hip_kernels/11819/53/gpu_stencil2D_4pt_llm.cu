#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_DIM_X blockDim.x * TILE_SIZE
#define TILE_DIM_Y blockDim.y * TILE_SIZE

__global__ void gpu_stencil2D_4pt(double *dst, double *src, int M, int N) {
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double *shSrc = shared_mem;

    int smColDim = HALO*2 + TILE_DIM_Y; // SharedMem Columns Dimension
    int smRowDim = HALO*2 + TILE_DIM_X; // SharedMem Rows Dimension

    // Calculate the indices only once for performance improvement
    int baseGlobalIdx = blockIdx.x * TILE_DIM_X * N + blockIdx.y * TILE_DIM_Y;
    int threadGlobalIdx = baseGlobalIdx + threadIdx.x * TILE_SIZE * N + threadIdx.y * TILE_SIZE; 

    int baseShMemIdx = HALO * smColDim + HALO;
    int threadShMemIdx = baseShMemIdx + threadIdx.x * smColDim * TILE_SIZE + threadIdx.y * TILE_SIZE;

    // Copying to shared memory
    // Inner part
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = threadGlobalIdx + i * N + j + HALO * N + HALO;
            int shMemIndex = threadShMemIdx + i * smColDim + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Halo regions copied by a single thread per block
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smColDim; j++) {
                int indexTopHalo = (blockIdx.x * TILE_DIM_X + i) * N + blockIdx.y * TILE_DIM_Y + j;
                int indexBottomHalo = (HALO + (blockIdx.x + 1) * TILE_DIM_X) * N + blockIdx.y * TILE_DIM_Y + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + TILE_DIM_X + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smRowDim - HALO * 2; j++) {
                int indexLeftHalo = (HALO + blockIdx.x * TILE_DIM_X + j) * N + blockIdx.y * TILE_DIM_Y + i;
                int indexRightHalo = (HALO + blockIdx.x * TILE_DIM_X + j) * N + (blockIdx.y + 1) * TILE_DIM_Y + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Stencil operation
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = threadGlobalIdx + i * N + j + HALO * N + HALO;
            int shMemIndex = threadShMemIdx + i * smColDim + j;

            // Getting the neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east = shSrc[shMemIndex + 1];
            double west = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }

    __syncthreads();
}