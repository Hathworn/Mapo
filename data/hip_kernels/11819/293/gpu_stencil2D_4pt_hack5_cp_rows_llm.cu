#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt_hack5_cp_rows(double * dst, double * shared_cols, double *shared_rows,int tile_y,int M, int N){

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
printf("copy rows begin!\n");
}
#endif

int base_global_row = tile_y * blockIdx.y;
int base_global_col = blockDim.x * blockIdx.x;
int base_global_idx = N * base_global_row + base_global_col;
int nextRow = base_global_row + 1;
bool legalNextRow = nextRow < M;
int t = threadIdx.x;
bool legalCurCol = base_global_col + t < N;
int shared_idx = (base_global_row / tile_y) * 2 * N + t + base_global_col;
int shared_idx_nextrow = shared_idx + N;

// Use of conditional stores to avoid warp divergence
if (legalCurCol) {
    shared_rows[shared_idx] = dst[base_global_idx + t];
    if (legalNextRow) {
        shared_rows[shared_idx_nextrow] = dst[base_global_idx + N + t];
    }
}
__syncthreads();

#ifdef CUDA_CUDA_DEBUG
if(blockIdx.y==0 && blockIdx.x==2 && (t==0 || t==1)){
printf("addr:%d, val = %f\n", shared_idx_nextrow, shared_rows[shared_idx_nextrow]);
}
#endif

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
printf("copy rows finish!\n");
}
#endif
}