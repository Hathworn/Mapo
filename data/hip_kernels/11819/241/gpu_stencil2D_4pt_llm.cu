#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declare shared memory for source
    extern __shared__ double shared_mem[];
    double *shSrc = shared_mem;

    // Local thread coordinates
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Shared memory dimensions
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Coordinates for global memory access
    int baseGlobalIndex = HALO * N + blockIdx.x * blockDim.x * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE;
    int shMemBaseIndex = HALO * smColDim + tx * smColDim * TILE_SIZE + HALO + ty * TILE_SIZE;

    // Copy inner part to shared memory
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = baseGlobalIndex + tx * TILE_SIZE * N + i * N + ty * TILE_SIZE + j;
            int shMemIndex = shMemBaseIndex + i * smColDim + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Copy halos to shared memory (only by thread (0, 0) to avoid redundancy)
    if (tx == 0 && ty == 0)
    {
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smColDim; j++)
            {
                int indexTopHalo = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                int indexBottomHalo = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smRowDim - HALO * 2; j++)
            {
                int indexLeftHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + blockIdx.y * blockDim.y * TILE_SIZE + i;
                int indexRightHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Compute stencil operation
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = baseGlobalIndex + tx * TILE_SIZE * N + i * N + ty * TILE_SIZE + j;
            int shMemIndex = shMemBaseIndex + i * smColDim + j;

            // Get neighboring values
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];

            // Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }

    __syncthreads();
}