#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil37_hack1_cp_cols(double * dst, double * shared_rows, double *shared_cols, double *shared_slices, int n_rows, int n_cols, int n_slices, int tile_x, int tile_y, int tile_z) {
    // Define base indices and area calculations
    int base_global_slice = tile_z * blockIdx.z;
    int base_global_row = blockDim.y * blockIdx.y;
    int base_global_col = tile_x * blockIdx.x;

    int area_dst = n_rows * n_cols;
    int area_shared = gridDim.x * n_rows * 2;

    int base_global_idx = base_global_slice * area_dst + base_global_row * n_cols + base_global_col;

    int nextCol = base_global_col + 1;
    bool legalNextCol = (nextCol < n_cols);

    int ty = threadIdx.y;
    bool legalCurRow = (base_global_row + ty) < n_rows;

    // Loop over z-dimension tiles
    for (int tz = 0; tz < tile_z; ++tz) {
        bool legalCurSlice = (base_global_slice + tz) < n_slices;
        int idx_dst = base_global_idx + tz * area_dst + ty * n_cols;
        int idx = (base_global_slice + tz) * area_shared + blockIdx.x * 2 * n_rows + blockIdx.y * blockDim.y + ty;

        // Use conditional read and store for both current and next column
        if (legalCurRow && legalCurSlice) {
            shared_cols[idx] = dst[idx_dst];
            if (legalNextCol) {
                shared_cols[idx + n_rows] = dst[idx_dst + 1];
            }
        }

        __syncthreads(); // Synchronize threads within the block after each iteration
    }
    __syncthreads(); // Final synchronization to ensure completion of all operations
}