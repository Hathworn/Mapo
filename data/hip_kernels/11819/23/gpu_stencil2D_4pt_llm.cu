#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // Neighbor's values
    double north, south, east, west;

    // SharedMem Dimensions
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;
    
    int baseColIndex = HALO + blockIdx.x * blockDim.x * TILE_SIZE * N + threadIdx.x * TILE_SIZE * N;
    int baseRowIndex = HALO + blockIdx.y * blockDim.y * TILE_SIZE + threadIdx.y * TILE_SIZE;

    // Copying to shared memory (Inner part)
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = baseColIndex + i * N + baseRowIndex + j;
            int shMemIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + HALO + threadIdx.y * TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Halos (only copy once when both threads are zero)
    if (threadIdx.x == 0 && threadIdx.y == 0 )
    {
        int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;

        // Top and bottom rows
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smColDim; j++)
            {
                indexTopHalo = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                indexBottomHalo = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        // Right and left columns
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smRowDim - HALO * 2; j++)
            {
                indexLeftHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + i;
                indexRightHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Compute stencil
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = baseColIndex + i * N + baseRowIndex + j;
            int shMemIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + HALO + threadIdx.y * TILE_SIZE + j;

            // Getting the neighbors
            north = shSrc[shMemIndex - smColDim];
            south = shSrc[shMemIndex + smColDim];
            east  = shSrc[shMemIndex + 1];
            west  = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}