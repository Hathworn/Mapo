#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // SharedMem Collumns Dimension
    int smColDim = HALO*2 + blockDim.y*TILE_SIZE;
    int smRowDim = HALO*2 + blockDim.x*TILE_SIZE;

    // Global Index Calculations
    int globalBaseIndex = HALO*N + blockIdx.x*blockDim.x*TILE_SIZE*N + blockIdx.y*blockDim.y*TILE_SIZE + HALO;
    int localRowIndex = HALO*smColDim + threadIdx.x*smColDim*TILE_SIZE;
    int localColIndex = HALO + threadIdx.y*TILE_SIZE;

    // Optimized copying to shared memory
    for (int i = 0; i < TILE_SIZE; i++)
    {
        int rowBaseIdx = localRowIndex + i*smColDim;
        int globalOffset = globalBaseIndex + threadIdx.x*TILE_SIZE*N + i*N;

        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = globalOffset + threadIdx.y*TILE_SIZE + j;
            int shMemIndex = rowBaseIdx + localColIndex + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Shared threads sync for halo copying 
    __syncthreads();

    // Efficient halo copying using conditionals inside shared memory
    if (threadIdx.x == 0 && threadIdx.y == 0)
    {
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smColDim; j++)
            {
                int indexTopHalo = (blockIdx.x*blockDim.x*TILE_SIZE + i) * N + (blockIdx.y*blockDim.y*TILE_SIZE) + j;
                int indexBottomHalo = (HALO + (blockIdx.x+1)*blockDim.x*TILE_SIZE)*N + (blockIdx.y*blockDim.y*TILE_SIZE) + j;

                shSrc[i*smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x*TILE_SIZE + i)*smColDim + j] = src[indexBottomHalo];
            }
        }

        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smRowDim - HALO*2; j++)
            {
                int indexLeftHalo = (HALO + blockIdx.x*blockDim.x*TILE_SIZE + j) * N + (blockIdx.y*blockDim.y*TILE_SIZE) + i;
                int indexRightHalo = (HALO + blockIdx.x*blockDim.x*TILE_SIZE + j) * N + ((blockIdx.y+1)*blockDim.y*TILE_SIZE) + HALO + i;

                shSrc[(HALO + j)*smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1)*smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    // Shared threads sync before stencil calculations
    __syncthreads();

    // Stencil calculation using shared memory
    for (int i = 0; i < TILE_SIZE; i++)
    {
        int rowBaseIdx = localRowIndex + i*smColDim;
        int globalOffset = globalBaseIndex + threadIdx.x*TILE_SIZE*N + i*N;

        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = globalOffset + threadIdx.y*TILE_SIZE + j;
            int shMemIndex = rowBaseIdx + localColIndex + j;

            // Getting the neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east = shSrc[shMemIndex + 1];
            double west = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }

    // Shared threads sync before finishing kernel
    __syncthreads();
}