#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt_hack5_cp_rows(double * dst, double * shared_cols, double *shared_rows, int tile_y, int M, int N) {

#ifdef CUDA_DARTS_DEBUG
    if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0)) {
        printf("copy rows begin!\n");
    }
#endif

    int base_global_row = tile_y * blockIdx.y;
    int base_global_col = blockDim.x * blockIdx.x;
    int base_global_idx = N * base_global_row + base_global_col;
    int nextRow = base_global_row + 1;
    int t = threadIdx.x;
    int idx = (base_global_row / tile_y) * 2 * N + t + base_global_col;

    // Optimized condition check using memory coalescing
    if ((base_global_col + t) < N) {
        shared_rows[idx] = dst[base_global_idx + t];
        if (nextRow < M) {
            shared_rows[idx + N] = dst[base_global_idx + N + t];
        }
    }

    __syncthreads();

#ifdef CUDA_CUDA_DEBUG
    if (blockIdx.y == 0 && blockIdx.x == 2 && (t == 0 || t == 1)) {
        printf("addr:%d, val = %f\n", idx + N, shared_rows[idx + N]);
    }
#endif

#ifdef CUDA_DARTS_DEBUG
    if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0)) {
        printf("copy rows finish!\n");
    }
#endif
}