#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // Simplify index calculations by using direct index variables
    int tx = threadIdx.x, ty = threadIdx.y;
    int blockIdx_x = blockIdx.x, blockIdx_y = blockIdx.y;
    int blockDim_x = blockDim.x, blockDim_y = blockDim.y;
    int tileIdxX = blockIdx_x * blockDim_x * TILE_SIZE;
    int tileIdxY = blockIdx_y * blockDim_y * TILE_SIZE;
    int smColDim = HALO * 2 + blockDim_y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim_x * TILE_SIZE;
    
    // Copying to shared memory - Inner part with loop unrolling
    for (int i = 0; i < TILE_SIZE; ++i) {
        for (int j = 0; j < TILE_SIZE; ++j) {
            int globalIndex = HALO * N + tileIdxX * N + tx * TILE_SIZE * N + i * N + tileIdxY + ty * TILE_SIZE + j + HALO;
            int shMemIndex = HALO * smColDim + tx * smColDim * TILE_SIZE + i * smColDim + HALO + ty * TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Load halos using specific threads only
    if (tx == 0 && ty == 0) {
        for (int i = 0; i < HALO; ++i) {
            for (int j = 0; j < smColDim; ++j) {
                // Top and Bottom Halos
                int indexTopHalo = (blockIdx_x * blockDim_x * TILE_SIZE + i) * N + tileIdxY + j;
                int indexBottomHalo = (HALO + (blockIdx_x + 1) * blockDim_x * TILE_SIZE) * N + tileIdxY + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim_x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }
        for (int i = 0; i < HALO; ++i) {
            for (int j = 0; j < smRowDim - HALO * 2; ++j) {
                // Left and Right Halos
                int indexLeftHalo = (HALO + blockIdx_x * blockDim_x * TILE_SIZE + j) * N + tileIdxY + i;
                int indexRightHalo = (HALO + blockIdx_x * blockDim_x * TILE_SIZE + j) * N + ((blockIdx_y + 1) * blockDim_y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Perform stencil operation with index recalculation
    for (int i = 0; i < TILE_SIZE; ++i) {
        for (int j = 0; j < TILE_SIZE; ++j) {
            int globalIndex = HALO * N + tileIdxX * N + tx * TILE_SIZE * N + i * N + tileIdxY + ty * TILE_SIZE + j + HALO;
            int shMemIndex = HALO * smColDim + tx * smColDim * TILE_SIZE + i * smColDim + HALO + ty * TILE_SIZE + j;

            // Getting the neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east = shSrc[shMemIndex + 1];
            double west = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }

    __syncthreads();
}