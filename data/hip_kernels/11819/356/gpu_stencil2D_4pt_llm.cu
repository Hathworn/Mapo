#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declare the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // SharedMem Columns and Rows Dimension
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Calculate global and shared indices once
    int globalBaseIndex = HALO * N + blockIdx.x * blockDim.x * TILE_SIZE * N + threadIdx.x * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE + threadIdx.y * TILE_SIZE + HALO;
    int shMemBaseIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + HALO + threadIdx.y * TILE_SIZE;

    // Copying to shared memory
    for (int i = 0; i < TILE_SIZE; ++i) {
        for (int j = 0; j < TILE_SIZE; ++j) {
            shSrc[shMemBaseIndex + i * smColDim + j] = src[globalBaseIndex + i * N + j];
        }
    }

    // Load Halos only once to reduce overhead
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        for (int i = 0; i < HALO; ++i) {
            for (int j = 0; j < smColDim; ++j) {
                shSrc[i * smColDim + j] = src[(blockIdx.x * blockDim.x * TILE_SIZE + i) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[(HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j];
            }
        }

        for (int i = 0; i < HALO; ++i) {
            for (int j = 0; j < smRowDim - HALO * 2; ++j) {
                shSrc[(HALO + j) * smColDim + i] = src[(HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + i];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[(HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i];
            }
        }
    }

    __syncthreads();

    for (int i = 0; i < TILE_SIZE; ++i) {
        for (int j = 0; j < TILE_SIZE; ++j) {
            int shMemIndex = shMemBaseIndex + i * smColDim + j;

            // Get the neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalBaseIndex + i * N + j] = (north + south + east + west) / 5.5;
        }
    }
    __syncthreads();
}