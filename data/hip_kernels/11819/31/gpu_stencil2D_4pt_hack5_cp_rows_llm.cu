#include "hip/hip_runtime.h"
```c++
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt_hack5_cp_rows(double * dst, double * shared_cols, double *shared_rows,int tile_y,int M, int N){

// Debug print start of copy
#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
    printf("copy rows begin!\n");
}
#endif

// Calculate base indices and conditions
int base_global_row = tile_y * blockIdx.y;
int base_global_col = blockDim.x * blockIdx.x;
int base_global_idx = N * base_global_row + base_global_col;
int t = threadIdx.x;
bool legalCurCol = (base_global_col + t) < N;
int idx = (base_global_row/tile_y) * 2 * N + base_global_col + t;
int idx_nextrow = idx + N;

// Copy current row
if (legalCurCol) {
    shared_rows[idx] = dst[base_global_idx + t];
}

// Copy next row if within bounds
int nextRow = base_global_row + 1;
if (nextRow < M && legalCurCol) {
    shared_rows[idx_nextrow] = dst[base_global_idx + N + t];
}

__syncthreads();

// Debug print specific conditions
#ifdef CUDA_CUDA_DEBUG
if (blockIdx.y == 0 && blockIdx.x == 2 && (t == 0 || t == 1)) {
    printf("addr:%d, val = %f\n", idx_nextrow, shared_rows[idx_nextrow]);
}
#endif

// Debug print end of copy
#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(threadIdx.x==0)){
    printf("copy rows finish!\n");
}
#endif
}