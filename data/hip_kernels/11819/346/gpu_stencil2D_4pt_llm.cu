#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // Compute indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Compute global and shared memory indices for tiles
    int globalBaseIndex = HALO * N + bx * blockDim.x * TILE_SIZE * N + tx * TILE_SIZE * N + by * blockDim.y * TILE_SIZE + ty * TILE_SIZE + HALO;
    int shMemBaseIndex = HALO * smColDim + tx * smColDim * TILE_SIZE + HALO + ty * TILE_SIZE;

    // Load tile into shared memory
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = globalBaseIndex + i * N + j;
            int shMemIndex = shMemBaseIndex + i * smColDim + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Synchronize to make sure the data is loaded
    __syncthreads();

    // Load halo regions only once
    if (tx == 0 && ty == 0)
    {
        // Top and bottom halo
        for (int j = 0; j < smColDim; j++) {
            shSrc[j] = src[(bx * blockDim.x * TILE_SIZE) * N + (by * blockDim.y * TILE_SIZE) + j];
            shSrc[(HALO + blockDim.x * TILE_SIZE) * smColDim + j] = src[(HALO + (bx + 1) * blockDim.x * TILE_SIZE) * N + (by * blockDim.y * TILE_SIZE) + j];
        }
        // Left and right halo
        for (int i = 0; i < smRowDim - HALO * 2; i++) {
            shSrc[(HALO + i) * smColDim] = src[(HALO + bx * blockDim.x * TILE_SIZE + i) * N + (by * blockDim.y * TILE_SIZE)];
            shSrc[(HALO + i + 1) * smColDim - HALO] = src[(HALO + bx * blockDim.x * TILE_SIZE + i) * N + ((by + 1) * blockDim.y * TILE_SIZE) + HALO];
        }
    }

    // Synchronize to make sure the edges are loaded
    __syncthreads();

    // Stencil computation
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = globalBaseIndex + i * N + j;
            int shMemIndex = shMemBaseIndex + i * smColDim + j;
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}