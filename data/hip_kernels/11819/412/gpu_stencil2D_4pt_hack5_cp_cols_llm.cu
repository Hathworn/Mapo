#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt_hack5_cp_cols(double * dst, double * shared_cols, double * shared_rows, int tile_x, int tile_y, int M, int N) {

#ifdef CUDA_DARTS_DEBUG
    if((blockIdx.x==0) && (blockIdx.y==0) && (threadIdx.y==0)){
        printf("copy cols begin!\n");
    }
#endif

    int base_global_row = tile_y * blockIdx.y;
    int base_global_col = tile_x * blockIdx.x;
    int base_global_idx = N * base_global_row + base_global_col;
    int t = threadIdx.y;
    int global_row = base_global_row + t;
    int idx = 2 * M * blockIdx.x + t + base_global_row;

    // Check boundary conditions only once
    if(global_row < M) {
        shared_cols[idx] = dst[base_global_idx + t * N]; // Load current column
        int nextColIdx = idx + M; // M is the stride for the next column

        if((base_global_col + 1) < N) // Check if next column is within bounds
            shared_cols[nextColIdx] = dst[base_global_idx + t * N + 1]; // Load next column
    }
    
    __syncthreads();

#ifdef CUDA_CUDA_DEBUG
    if(blockIdx.x == 1 && t < 5){
        printf("addr: %d ,%f,\n", idx + M, shared_cols[idx + M]);
    }
#endif

#ifdef CUDA_DARTS_DEBUG
    if((blockIdx.x==0) && (blockIdx.y==0) && (threadIdx.y==0)){
        printf("copy cols finish!\n");
    }
#endif
}