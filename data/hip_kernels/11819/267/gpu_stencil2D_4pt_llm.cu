#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Use shared memory for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // Calculate shared memory dimensions
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Compute global start index
    int global_i = blockIdx.x * blockDim.x * TILE_SIZE + HALO + threadIdx.x * TILE_SIZE;
    int global_j = blockIdx.y * blockDim.y * TILE_SIZE + HALO + threadIdx.y * TILE_SIZE;

    // Load inner tile to shared memory
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = (global_i + i) * N + (global_j + j);
            int shMemIndex = (threadIdx.x * TILE_SIZE + i + HALO) * smColDim + (threadIdx.y * TILE_SIZE + j + HALO);
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Synchronize to ensure all threads have loaded the data
    __syncthreads();

    // Load halo elements only if using first thread
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        for (int i = 0; i < HALO; i++) {
            // Load top and bottom halos
            for (int j = 0; j < smColDim; j++) {
                int indexTop = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                int indexBottom = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                shSrc[i * smColDim + j] = src[indexTop];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottom];
            }

            // Load left and right halos
            for (int j = 0; j < smRowDim - HALO * 2; j++) {
                int indexLeft = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + i;
                int indexRight = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeft];
                shSrc[(HALO + j) * smColDim + smColDim - HALO + i] = src[indexRight];
            }
        }
    }

    // Synchronize again before stencil computation
    __syncthreads();

    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = (global_i + i) * N + (global_j + j);
            int shMemIndex = (threadIdx.x * TILE_SIZE + i + HALO) * smColDim + (threadIdx.y * TILE_SIZE + j + HALO);

            // Perform stencil operation
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}