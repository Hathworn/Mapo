#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // Calculate shared memory dimensions
    int smColDim = HALO*2+blockDim.y*TILE_SIZE;
    int smRowDim = HALO*2+blockDim.x*TILE_SIZE;

    // Calculate global indices
    int globalStartX = blockIdx.x*blockDim.x*TILE_SIZE + threadIdx.x*TILE_SIZE;
    int globalStartY = blockIdx.y*blockDim.y*TILE_SIZE + threadIdx.y*TILE_SIZE;

    // Load inner part into shared memory
    for (int i = 0; i < TILE_SIZE; i++) 
    {
        for (int j = 0; j < TILE_SIZE; j++) 
        {
            int globalIndex = HALO*N + globalStartX*N + i*N + globalStartY + j + HALO;
            int shMemIndex = HALO*smColDim + threadIdx.x*smColDim*TILE_SIZE + i*smColDim + HALO + threadIdx.y*TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }
    __syncthreads();

    // Load halos into shared memory
    if (threadIdx.x == 0 && threadIdx.y == 0) 
    {
        for (int i = 0; i < HALO; i++) 
        {
            for (int j = 0; j < smColDim; j++) 
            {
                int indexTopHalo = (blockIdx.x*blockDim.x*TILE_SIZE + i)*N + (blockIdx.y*blockDim.y*TILE_SIZE) + j;
                int indexBottomHalo = (HALO + (blockIdx.x+1)*blockDim.x*TILE_SIZE)*N + (blockIdx.y*blockDim.y*TILE_SIZE) + j;
                shSrc[i*smColDim+j] = src[indexTopHalo];
                shSrc[(HALO+blockDim.x*TILE_SIZE+i)*smColDim + j] = src[indexBottomHalo];
            }
        }
        
        for (int i = 0; i < HALO; i++) 
        {
            for (int j = 0; j < smRowDim-HALO*2; j++) 
            {
                int indexLeftHalo = (HALO+blockIdx.x*blockDim.x*TILE_SIZE+j)*N + (blockIdx.y*blockDim.y*TILE_SIZE) + i;
                int indexRightHalo = (HALO+blockIdx.x*blockDim.x*TILE_SIZE+j)*N + ((blockIdx.y+1)*blockDim.y*TILE_SIZE) + HALO + i;
                shSrc[(HALO+j)*smColDim+i] = src[indexLeftHalo];
                shSrc[(HALO+j+1)*smColDim-HALO+i] = src[indexRightHalo];
            }
        }
    }
    __syncthreads();

    // Perform stencil operation
    for (int i = 0; i < TILE_SIZE; i++) 
    {
        for (int j = 0; j < TILE_SIZE; j++) 
        {
            int globalIndex = HALO*N + globalStartX*N + i*N + globalStartY + j + HALO;
            int shMemIndex = HALO*smColDim + threadIdx.x*smColDim*TILE_SIZE + i*smColDim + HALO + threadIdx.y*TILE_SIZE + j;

            // Read neighbors
            double north = shSrc[shMemIndex-smColDim];
            double south = shSrc[shMemIndex+smColDim];
            double east  = shSrc[shMemIndex+1];
            double west  = shSrc[shMemIndex-1];

            // Real stencil operation
            dst[globalIndex] = (north + south + east + west)/5.5;
        }
    }
}