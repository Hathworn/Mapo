#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // Precompute indices outside of the loops for reuse
    const int blockOffsetX = blockIdx.x * blockDim.x * TILE_SIZE;
    const int blockOffsetY = blockIdx.y * blockDim.y * TILE_SIZE;
    const int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    const int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;
  
    const int threadOffsetX = threadIdx.x * TILE_SIZE;
    const int threadOffsetY = threadIdx.y * TILE_SIZE;
  
    // Copying to shared memory
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = HALO * N + blockOffsetX * N + threadOffsetX * N + i * N + blockOffsetY + threadOffsetY + j + HALO;
            int shMemIndex = HALO * smColDim + threadOffsetX * smColDim + i * smColDim + HALO + threadOffsetY + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Halos, handled by one thread
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smColDim; j++) {
                int indexTopHalo = (blockOffsetX + i) * N + blockOffsetY + j;
                int indexBottomHalo = (HALO + blockOffsetX + blockDim.x * TILE_SIZE) * N + blockOffsetY + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        for (int j = 0; j < smRowDim - HALO * 2; j++) {
            for (int i = 0; i < HALO; i++) {
                int indexLeftHalo = (HALO + blockOffsetX + j) * N + blockOffsetY + i;
                int indexRightHalo = (HALO + blockOffsetX + j) * N + (blockOffsetY + blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }
    __syncthreads();

    // Stencil computation
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = HALO * N + blockOffsetX * N + threadOffsetX * N + i * N + blockOffsetY + threadOffsetY + j + HALO;
            int shMemIndex = HALO * smColDim + threadOffsetX * smColDim + i * smColDim + HALO + threadOffsetY + j;

            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];
  
            // Optimized stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
    __syncthreads();
}