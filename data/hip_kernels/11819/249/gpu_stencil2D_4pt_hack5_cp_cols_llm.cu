#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt_hack5_cp_cols(double * dst, double * shared_cols, double *shared_rows, int tile_x, int tile_y, int M, int N) {

#ifdef CUDA_DARTS_DEBUG
    if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.y == 0)) {
        printf("copy cols begin!\n");
    }
#endif

    int base_global_row = tile_y * blockIdx.y;
    int base_global_col = tile_x * blockIdx.x;
    int base_global_idx = N * base_global_row + base_global_col;
    int t = threadIdx.y;
    int idx = 2 * M * blockIdx.x + t + base_global_row;
    
    // Precompute only once
    int tN = t * N;
    int nextColOffset = M;
    
    if ((base_global_row + t) < M) {
        shared_cols[idx] = dst[base_global_idx + tN];
        
        // Combine the index calculations for nextCol
        if ((base_global_col + 1) < N) {
            shared_cols[idx + nextColOffset] = dst[base_global_idx + tN + 1];
        }
    }
    
    __syncthreads();

#ifdef CUDA_CUDA_DEBUG
    if (blockIdx.x == 1 && t < 5) {
        printf("addr: %d ,%f,\n", idx + nextColOffset, shared_cols[idx + nextColOffset]);
    }
#endif

#ifdef CUDA_DARTS_DEBUG
    if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.y == 0)) {
        printf("copy cols finish!\n");
    }
#endif
}