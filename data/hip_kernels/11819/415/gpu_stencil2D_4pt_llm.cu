#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // SharedMem Column Dimension
    int smColDim = HALO*2 + blockDim.y*TILE_SIZE;
    int smRowDim = HALO*2 + blockDim.x*TILE_SIZE;

    // Calculate blockIndices for halo loading only once
    int blockXOffset = blockIdx.x*blockDim.x*TILE_SIZE;
    int blockYOffset = blockIdx.y*blockDim.y*TILE_SIZE;

    // Copying to shared memory - Inner part
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = HALO*N + blockXOffset*N + tx*TILE_SIZE*N + i*N + blockYOffset + ty*TILE_SIZE + j + HALO;
            int shMemIndex = HALO*smColDim + tx*smColDim*TILE_SIZE + i*smColDim + HALO + ty*TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Synchronizing threads before halo copy
    __syncthreads();

    // Load halo only if necessary
    if (tx == 0 && ty == 0)
    {
        int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;
        
        // Halo loading - Top and Bottom rows
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smColDim; j++)
            {
                indexTopHalo = (blockXOffset + i)*N + blockYOffset + j;
                indexBottomHalo = (HALO + blockXOffset + blockDim.x*TILE_SIZE)*N + blockYOffset + j;
                shSrc[i*smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x*TILE_SIZE + i)*smColDim + j] = src[indexBottomHalo];
            }
        }

        // Halo loading - Left and Right Columns
        for (int i = 0; i < HALO; i++)
        {
            for (int j = 0; j < smRowDim - HALO*2; j++)
            {
                indexLeftHalo = (HALO + blockXOffset + j)*N + blockYOffset + i;
                indexRightHalo = (HALO + blockXOffset + j)*N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j)*smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1)*smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    // Synchronizing threads after halo copy
    __syncthreads();

    // Stencil operation on shared memory
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = HALO*N + blockXOffset*N + tx*TILE_SIZE*N + i*N + blockYOffset + ty*TILE_SIZE + j + HALO;
            int shMemIndex = HALO*smColDim + tx*smColDim*TILE_SIZE + i*smColDim + HALO + ty*TILE_SIZE + j;

            // Getting the neighbors
            north = shSrc[shMemIndex - smColDim];
            south = shSrc[shMemIndex + smColDim];
            east  = shSrc[shMemIndex + 1];
            west  = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}