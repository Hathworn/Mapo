#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declare shared memory
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // Determine shared memory dimensions
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Calculate global indices for each thread
    int baseI = HALO * N + blockIdx.x * blockDim.x * TILE_SIZE * N + threadIdx.x * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE + threadIdx.y * TILE_SIZE + HALO;
    int baseShIndex = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + HALO + threadIdx.y * TILE_SIZE;

    // Load the inner tile into shared memory
    for (int i = 0; i < TILE_SIZE; ++i) 
    {
        for (int j = 0; j < TILE_SIZE; ++j) 
        {
            int globalIndex = baseI + i * N + j;
            int shMemIndex = baseShIndex + i * smColDim + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Synchronize to ensure complete tile loading
    __syncthreads();

    // Load halo only if one thread performs this operation
    if (threadIdx.x == 0 && threadIdx.y == 0) 
    {
        int index, haloIndex;

        // Top and bottom halos
        for (int i = 0; i < HALO; ++i) 
        {
            for (int j = 0; j < smColDim; ++j) 
            {
                index = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                haloIndex = i * smColDim + j;
                shSrc[haloIndex] = src[index];

                index = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                haloIndex = (HALO + blockDim.x * TILE_SIZE + i) * smColDim + j;
                shSrc[haloIndex] = src[index];
            }
        }

        // Left and right halos
        for (int j = 0; j < smRowDim - HALO * 2; ++j) 
        {
            for (int i = 0; i < HALO; ++i) 
            {
                index = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + blockIdx.y * blockDim.y * TILE_SIZE + i;
                haloIndex = (HALO + j) * smColDim + i;
                shSrc[haloIndex] = src[index];

                index = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                haloIndex = (HALO + j + 1) * smColDim - HALO + i;
                shSrc[haloIndex] = src[index];
            }
        }
    }

    // Synchronize to ensure halo loading
    __syncthreads();

    // Perform stencil computation
    for (int i = 0; i < TILE_SIZE; ++i) 
    {
        for (int j = 0; j < TILE_SIZE; ++j) 
        {
            int globalIndex = baseI + i * N + j;
            int shMemIndex = baseShIndex + i * smColDim + j;

            // Retrieve neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east = shSrc[shMemIndex + 1];
            double west = shSrc[shMemIndex - 1];

            // Apply stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }

    // Synchronize at end of kernel
    __syncthreads();
}