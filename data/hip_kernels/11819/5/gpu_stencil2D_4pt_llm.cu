#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}
#define ROTATE_DOWN(val,MAX) ((val-1==-1)?MAX-1:val-1)
#define ROTATE_UP(val,MAX) ((val+1)%MAX)

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declare shared memory for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // Determine block indices and size constants
    int smColDim = HALO*2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO*2 + blockDim.x * TILE_SIZE;
    int tileBaseRow = blockIdx.x * blockDim.x * TILE_SIZE;
    int tileBaseCol = blockIdx.y * blockDim.y * TILE_SIZE;

    // Efficiently copy to shared memory using one-dimensional thread index
    int tIdx = threadIdx.y * blockDim.x + threadIdx.x;
    int tDim = blockDim.x * blockDim.y;

    for (int i = tIdx; i < TILE_SIZE * TILE_SIZE; i += tDim) {
        int localRow = i / TILE_SIZE;
        int localCol = i % TILE_SIZE;
        int globalIndex = HALO * N + tileBaseRow * N + localRow * N + tileBaseCol + HALO + localCol;
        int shMemIndex = HALO * smColDim + localRow * smColDim + HALO + localCol;
        shSrc[shMemIndex] = src[globalIndex];
    }

    __syncthreads();

    // Load halos for top/bottom and left/right borders
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        for (int i = 0; i < HALO; ++i) {
            for (int j = 0; j < smColDim; ++j) {
                // Top halo
                shSrc[i * smColDim + j] = src[(tileBaseRow + i) * N + tileBaseCol + j];
                // Bottom halo
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[(HALO + tileBaseRow + blockDim.x * TILE_SIZE + i) * N + tileBaseCol + j];
            }
            for (int j = HALO; j < smRowDim - HALO; ++j) {
                // Left halo
                shSrc[j * smColDim + i] = src[(HALO + tileBaseRow + j - HALO) * N + tileBaseCol + i];
                // Right halo
                shSrc[j * smColDim + smColDim - HALO + i] = src[(HALO + tileBaseRow + j - HALO) * N + tileBaseCol + blockDim.y * TILE_SIZE + i];
            }
        }
    }

    __syncthreads();

    // Perform stencil operation
    for (int i = tIdx; i < TILE_SIZE * TILE_SIZE; i += tDim) {
        int localRow = i / TILE_SIZE;
        int localCol = i % TILE_SIZE;
        int globalIndex = HALO * N + tileBaseRow * N + localRow * N + tileBaseCol + HALO + localCol;
        int shMemIndex = HALO * smColDim + localRow * smColDim + HALO + localCol;
        
        double north = shSrc[shMemIndex - smColDim];
        double south = shSrc[shMemIndex + smColDim];
        double east  = shSrc[shMemIndex + 1];
        double west  = shSrc[shMemIndex - 1];
        
        // Apply stencil operation
        dst[globalIndex] = (north + south + east + west) / 5.5;
    }

    __syncthreads();
}