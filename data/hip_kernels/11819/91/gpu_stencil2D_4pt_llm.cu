#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declare shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // Calculate common indices once outside loops for efficiency
    int smColDim = HALO*2 + blockDim.y*TILE_SIZE;
    int smRowDim = HALO*2 + blockDim.x*TILE_SIZE;

    // Local thread indices
    int local_i = threadIdx.x;
    int local_j = threadIdx.y;

    // Compute global base index for this thread
    int base_global_index = HALO*N + blockIdx.x*blockDim.x*TILE_SIZE*N + blockIdx.y*blockDim.y*TILE_SIZE + local_i*TILE_SIZE*N + local_j*TILE_SIZE + HALO;
    int base_shMemIndex = HALO*smColDim + local_i*smColDim*TILE_SIZE + HALO + local_j*TILE_SIZE;

    // Copy input to shared memory (inner part)
    for (int i = 0; i < TILE_SIZE; i++) {
        #pragma unroll // Enable loop unrolling
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = base_global_index + i*N + j;
            int shMemIndex = base_shMemIndex + i*smColDim + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Handle halos
    if (local_i == 0 && local_j == 0)
    {
        int idx_top, idx_bottom, idx_left, idx_right;
        // Handle top and bottom halos
        for (int i = 0; i < HALO; i++) {
            #pragma unroll
            for (int j = 0; j < smColDim; j++) {
                idx_top = (blockIdx.x*blockDim.x*TILE_SIZE+i)*N + (blockIdx.y*blockDim.y*TILE_SIZE) + j;
                idx_bottom = (HALO + (blockIdx.x+1)*blockDim.x*TILE_SIZE)*N + (blockIdx.y*blockDim.y*TILE_SIZE) + j;
                shSrc[i*smColDim + j] = src[idx_top];
                shSrc[(HALO+blockDim.x*TILE_SIZE+i)*smColDim + j] = src[idx_bottom];
            }
        }
        // Handle left and right halos
        for (int i = 0; i < HALO; i++) {
            #pragma unroll
            for (int j = 0; j < smRowDim-HALO*2; j++) {
                idx_left = (HALO + blockIdx.x*blockDim.x*TILE_SIZE + j)*N + (blockIdx.y*blockDim.y*TILE_SIZE) + i;
                idx_right = (HALO + blockIdx.x*blockDim.x*TILE_SIZE + j)*N + ((blockIdx.y+1)*blockDim.y*TILE_SIZE) + HALO + i;
                shSrc[(HALO+j)*smColDim + i] = src[idx_left];
                shSrc[(HALO+j+1)*smColDim - HALO + i] = src[idx_right];
            }
        }
    }

    __syncthreads();

    // Apply stencil operation
    for (int i = 0; i < TILE_SIZE; i++) {
        #pragma unroll
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = base_global_index + i*N + j;
            int shMemIndex = base_shMemIndex + i*smColDim + j;

            // Read neighbor values
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];

            // Perform stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}