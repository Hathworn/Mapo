#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double* dst, double* src, int M, int N) {
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double* shSrc = shared_mem;

    // Compute thread indexes
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // SharedMem dimensions
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Compute global base index from block and thread index
    int baseGlobalIdx = HALO * N + bx * blockDim.x * TILE_SIZE * N + by * blockDim.y * TILE_SIZE;

    // Load inner data to shared memory
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = baseGlobalIdx + tx * TILE_SIZE * N + i * N + ty * TILE_SIZE + j + HALO;
            int shMemIndex = HALO * smColDim + tx * smColDim * TILE_SIZE + i * smColDim + HALO + ty * TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Load halo regions to shared memory
    if (tx == 0 && ty == 0) {
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smColDim; j++) {
                int indexTopHalo = bx * blockDim.x * TILE_SIZE * N + i * N + by * blockDim.y * TILE_SIZE + j;
                int indexBottomHalo = (HALO + (bx + 1) * blockDim.x * TILE_SIZE) * N + by * blockDim.y * TILE_SIZE + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smRowDim - HALO * 2; j++) {
                int indexLeftHalo = (HALO + bx * blockDim.x * TILE_SIZE + j) * N + by * blockDim.y * TILE_SIZE + i;
                int indexRightHalo = (HALO + bx * blockDim.x * TILE_SIZE + j) * N + ((by + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Perform stencil operation
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = baseGlobalIdx + tx * TILE_SIZE * N + i * N + ty * TILE_SIZE + j + HALO;
            int shMemIndex = HALO * smColDim + tx * smColDim * TILE_SIZE + i * smColDim + HALO + ty * TILE_SIZE + j;

            // Get neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east  = shSrc[shMemIndex + 1];
            double west  = shSrc[shMemIndex - 1];

            // Calculate stencil and apply the operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }

    __syncthreads();
}