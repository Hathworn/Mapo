#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double *dst, double *src, int M, int N) {
    // Declare shared memory
    extern __shared__ double shared_mem[];
    double *shSrc = shared_mem;

    // Initialize indexes
    int i, j;

    // Neighbor's values
    double north, south, east, west;

    // Define SharedMem Dimensions
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Compute global and shared memory indices
    int globalIndexBase = HALO * N + blockIdx.x * blockDim.x * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE;
    int shMemIndexBase = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE + HALO + threadIdx.y * TILE_SIZE;

    // Copy data to shared memory
    for (i = 0; i < TILE_SIZE; i++) {
        for (j = 0; j < TILE_SIZE; j++) {
            int globalIndex = globalIndexBase + threadIdx.x * TILE_SIZE * N + i * N + j + HALO;
            int shMemIndex = shMemIndexBase + i * smColDim + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Sync threads to ensure shared data is ready
    __syncthreads();

    // Handle halo regions using a single thread in each block
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        // Prepare indices for halo operations
        for (i = 0; i < HALO; i++) {
            for (j = 0; j < smColDim; j++) {
                // Top and bottom halos
                int indexTopHalo = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                int indexBottomHalo = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }
        for (j = 0; j < smRowDim - HALO * 2; j++) {
            // Left and right halos
            int indexLeftHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + blockIdx.y * blockDim.y * TILE_SIZE;
            int indexRightHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO;
            shSrc[(HALO + j) * smColDim] = src[indexLeftHalo];
            shSrc[(HALO + j + 1) * smColDim - HALO] = src[indexRightHalo];
        }
    }

    // Sync before reading shared data
    __syncthreads();

    // Compute stencil operation
    for (i = 0; i < TILE_SIZE; i++) {
        for (j = 0; j < TILE_SIZE; j++) {
            int globalIndex = globalIndexBase + threadIdx.x * TILE_SIZE * N + i * N + threadIdx.y * TILE_SIZE + j + HALO;
            int shMemIndex = shMemIndexBase + i * smColDim + j;

            // Fetch neighbors
            north = shSrc[shMemIndex - smColDim];
            south = shSrc[shMemIndex + smColDim];
            east  = shSrc[shMemIndex + 1];
            west  = shSrc[shMemIndex - 1];

            // Perform actual stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}