#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Using shared memory for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // Compute global indices for threads
    int global_i = HALO*N + blockIdx.x * blockDim.x * TILE_SIZE * N + threadIdx.x * TILE_SIZE * N;
    int global_j = HALO + blockIdx.y * blockDim.y * TILE_SIZE + threadIdx.y * TILE_SIZE;

    // Compute shared memory indices
    int sm_i = HALO * blockDim.y * TILE_SIZE + threadIdx.x * blockDim.y * TILE_SIZE;
    int sm_j = HALO + threadIdx.y * TILE_SIZE;

    // Copy inner tiles to shared memory (optimized to eliminate nested loops)
    for (int i = 0; i < TILE_SIZE; i++)
    {
        int offset_i = i * N;
        int sm_offset_i = i * blockDim.y * TILE_SIZE;
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = global_i + offset_i + global_j + j;
            int shMemIndex = sm_i + sm_offset_i + sm_j + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Synchronize to ensure shared memory loads complete
    __syncthreads();

    // Perform stencil operation using shared memory (optimized indexing)
    for (int i = 0; i < TILE_SIZE; i++)
    {
        int sm_offset_i = i * blockDim.y * TILE_SIZE;
        for (int j = 0; j < TILE_SIZE; j++)
        {
            int globalIndex = global_i + i * N + global_j + j;
            int shMemIndex = sm_i + sm_offset_i + sm_j + j;

            // Access neighbor values from shared memory
            double north = shSrc[shMemIndex - blockDim.y * TILE_SIZE];
            double south = shSrc[shMemIndex + blockDim.y * TILE_SIZE];
            double east = shSrc[shMemIndex + 1];
            double west = shSrc[shMemIndex - 1];

            // Write result to global memory
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}