#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // SharedMem Collumns Dimension
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Calculate global and shared memory indexes
    int globalBaseIndex = HALO * N + blockIdx.x * blockDim.x * TILE_SIZE * N + blockIdx.y * blockDim.y * TILE_SIZE + HALO;
    int shMemBaseIndex = HALO * smColDim + HALO;

    // Copying to shared memory (Inner part)
    for (int i = 0; i < TILE_SIZE; ++i) {
        for (int j = 0; j < TILE_SIZE; ++j) {
            int globalIndex = globalBaseIndex + threadIdx.x * TILE_SIZE * N + i * N + threadIdx.y * TILE_SIZE + j;
            int shMemIndex = shMemBaseIndex + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + threadIdx.y * TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Synchronize threads before adding halos
    __syncthreads();

    // Halos (performed by a single thread)
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        for (int i = 0; i < HALO; ++i) {
            for (int j = 0; j < smColDim; ++j) {
                int topHaloIndex = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                int bottomHaloIndex = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + blockIdx.y * blockDim.y * TILE_SIZE + j;
                shSrc[i * smColDim + j] = src[topHaloIndex];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[bottomHaloIndex];
            }
        }

        for (int i = 0; i < HALO; ++i) {
            for (int j = 0; j < smRowDim - HALO * 2; ++j) {
                int leftHaloIndex = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + blockIdx.y * blockDim.y * TILE_SIZE + i;
                int rightHaloIndex = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + (blockIdx.y + 1) * blockDim.y * TILE_SIZE + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[leftHaloIndex];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[rightHaloIndex];
            }
        }
    }

    // Synchronize threads before computing the stencil
    __syncthreads();

    // Stencil calculations
    for (int i = 0; i < TILE_SIZE; ++i) {
        for (int j = 0; j < TILE_SIZE; ++j) {
            int globalIndex = globalBaseIndex + threadIdx.x * TILE_SIZE * N + i * N + threadIdx.y * TILE_SIZE + j;
            int shMemIndex = shMemBaseIndex + threadIdx.x * smColDim * TILE_SIZE + i * smColDim + threadIdx.y * TILE_SIZE + j;

            // Getting the neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east = shSrc[shMemIndex + 1];
            double west = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}