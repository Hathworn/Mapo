#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N) {
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double * shSrc = shared_mem;

    // SharedMem Columns and Rows Dimension
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Calculate thread's global and shared coordinates
    int baseGlobalIndexX = HALO * N + blockIdx.x * blockDim.x * TILE_SIZE * N;
    int baseGlobalIndexY = blockIdx.y * blockDim.y * TILE_SIZE + HALO;
    int baseShMemIndexX = HALO * smColDim + threadIdx.x * smColDim * TILE_SIZE;
    int baseShMemIndexY = HALO + threadIdx.y * TILE_SIZE;

    // Load data into shared memory
    for (int i = 0; i < TILE_SIZE; i++) {
        #pragma unroll  // Unrolling loop for potential register reuse
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = baseGlobalIndexX + threadIdx.x * TILE_SIZE * N + i * N + baseGlobalIndexY + threadIdx.y * TILE_SIZE + j;
            int shMemIndex = baseShMemIndexX + i * smColDim + baseShMemIndexY + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Handle Halos using a single thread per block
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        int indexTopHalo, indexBottomHalo;
        int indexLeftHalo, indexRightHalo;

        // For Bottom and top row
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smColDim; j++) {
                indexTopHalo = (blockIdx.x * blockDim.x * TILE_SIZE + i) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                indexBottomHalo = (HALO + (blockIdx.x + 1) * blockDim.x * TILE_SIZE) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        // For right and left Columns
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smRowDim - HALO * 2; j++) {
                indexLeftHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + (blockIdx.y * blockDim.y * TILE_SIZE) + i;
                indexRightHalo = (HALO + blockIdx.x * blockDim.x * TILE_SIZE + j) * N + ((blockIdx.y + 1) * blockDim.y * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Apply stencil operation
    for (int i = 0; i < TILE_SIZE; i++) {
        #pragma unroll  // Unrolling loop for potential performance gains
        for (int j = 0; j < TILE_SIZE; j++) {
            int globalIndex = baseGlobalIndexX + threadIdx.x * TILE_SIZE * N + i * N + baseGlobalIndexY + threadIdx.y * TILE_SIZE + j;
            int shMemIndex = baseShMemIndexX + i * smColDim + baseShMemIndexY + j;

            // Getting the neighbors
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east = shSrc[shMemIndex + 1];
            double west = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }
}