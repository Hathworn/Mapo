#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt_hack5_cp_cols(double *dst, double *shared_cols, double *shared_rows, int tile_x, int tile_y, int M, int N){

#ifdef CUDA_DARTS_DEBUG
    if((blockIdx.x==0) && (blockIdx.y==0) && (threadIdx.y==0)) {
        printf("copy cols begin!\n");
    }
#endif

    int base_global_row = tile_y * blockIdx.y;
    int base_global_col = tile_x * blockIdx.x;
    int base_global_idx = N * base_global_row + base_global_col;
    bool legalCurRow = (base_global_row + threadIdx.y) < M;
    int nextCol = base_global_col + 1;
    bool legalNextCol = nextCol < N;
    int idx = 2 * M * blockIdx.x + threadIdx.y + base_global_row;
    
    if(legalCurRow) {
        int global_offset = base_global_idx + threadIdx.y * N; // Calculate global offset once
        shared_cols[idx] = dst[global_offset];
        if(legalNextCol) {
            shared_cols[idx + M] = dst[global_offset + 1];
        }
    }
    __syncthreads();

#ifdef CUDA_CUDA_DEBUG
    if(blockIdx.x == 1 && threadIdx.y < 5) {
        printf("addr: %d ,%f,\n", idx + M, shared_cols[idx + M]);
    }
#endif

#ifdef CUDA_DARTS_DEBUG
    if((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.y == 0)) {
        printf("copy cols finish!\n");
    }
#endif
}