#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil37_hack1_cp_slices(double * dst, double * shared_rows, double *shared_cols,double *shared_slices,int n_rows, int n_cols,int n_slices,int tile_x,int tile_y, int tile_z){

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
printf("copy slices begin!\n");
printf("gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n",gridDim.x,gridDim.y,gridDim.z);
printf("blockDim.x=%d,blockDim.y=%d,blockDim.z=%d\n",blockDim.x,blockDim.y,blockDim.z);
printf("tile_x=%d,tile_y=%d,tile_z=%d\n",tile_x,tile_y,tile_z);
}
#endif

int base_global_slice = tile_z * blockIdx.z;
int base_global_row = tile_y * blockIdx.y;
int base_global_col = blockDim.x * blockIdx.x;
int area = n_rows * n_cols;
int base_global_idx = base_global_slice * area + base_global_row * n_cols + base_global_col;
int nextSlice = base_global_slice + 1;
bool legalNextSlice = (nextSlice < n_slices);
int tx = threadIdx.x;
bool legalCurCol = (base_global_col + tx) < n_cols;

// Combine checks into the loop header
for (int ty = 0; ty < tile_y && (base_global_row + ty) < n_rows; ++ty) {
    int idx = blockIdx.z * area * 2 + (base_global_row + ty) * n_cols + base_global_col + tx;
    int idx_dst = base_global_idx + ty * n_cols + tx;
    if (legalCurCol) {
        shared_slices[idx] = dst[idx_dst];
        if (legalNextSlice) {
            shared_slices[idx + area] = dst[idx_dst + area];
        }
    }
}

__syncthreads();

#ifdef CUDA_CUDA_DEBUG
if(blockIdx.z ==0 && blockIdx.y==0 && blockIdx.x==1 ){
    if(threadIdx.x==0||threadIdx.x==1||threadIdx.x==2){
        int base = n_cols * n_rows + blockDim.x * blockIdx.x + threadIdx.x;
        printf("blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,shared_slices: addr= %d, val= %f\n", blockIdx.x, blockIdx.y, blockIdx.z, base, shared_slices[base]);
        printf("blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,shared_slices: addr= %d, val= %f\n", blockIdx.x, blockIdx.y, blockIdx.z, base + n_cols, shared_slices[base + n_cols]);
        printf("blockIdx.x=%d, blockIdx.y=%d, blockIdx.z=%d,shared_slices: addr= %d, val= %f\n", blockIdx.x, blockIdx.y, blockIdx.z, base + n_cols * 2, shared_slices[base + n_cols * 2]);
    }
}
#endif

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.x==0)){
printf("copy slices end!\n");
}
#endif
}