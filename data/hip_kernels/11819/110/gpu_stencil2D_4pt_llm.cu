#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil2D_4pt(double * dst, double * src, int M, int N)
{
    // Declaring the shared memory array for source
    extern __shared__ double shared_mem[];
    double *shSrc = shared_mem;

    // indexes
    int i, j;

    // neighbor's values
    double north, south, east, west;

    // SharedMem Columns Dimension
    int smColDim = HALO*2 + blockDim.y*TILE_SIZE;
    int smRowDim = HALO*2 + blockDim.x*TILE_SIZE;

    // Copying to shared memory
    int globalIndex, shMemIndex;

    // Copying the inner part
#pragma unroll
    for (i = 0; i < TILE_SIZE; i++) {
        for (j = 0; j < TILE_SIZE; j++) {
            globalIndex = (HALO + blockIdx.x*blockDim.x*TILE_SIZE + threadIdx.x*TILE_SIZE + i)*N 
                        + blockIdx.y*blockDim.y*TILE_SIZE + threadIdx.y*TILE_SIZE + j + HALO;
            shMemIndex = (HALO + threadIdx.x*TILE_SIZE + i)*smColDim 
                       + HALO + threadIdx.y*TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Handling halos in parallel using threads
    __syncthreads();
    if (threadIdx.x < HALO) {
        for (j = 0; j < smColDim; j++) {
            // Top halo
            int idxTopHalo = (blockIdx.x*blockDim.x*TILE_SIZE + threadIdx.x)*N 
                           + blockIdx.y*blockDim.y*TILE_SIZE + j;
            shSrc[threadIdx.x*smColDim + j] = src[idxTopHalo];

            // Bottom halo
            int idxBottomHalo = (HALO + (blockIdx.x + 1)*blockDim.x*TILE_SIZE + threadIdx.x)*N 
                              + blockIdx.y*blockDim.y*TILE_SIZE + j;
            shSrc[(HALO + blockDim.x*TILE_SIZE + threadIdx.x)*smColDim + j] = src[idxBottomHalo];
        }
    }

    if (threadIdx.y < HALO) {
        for (i = 0; i < smRowDim; i++) {
            // Left halo
            int idxLeftHalo = (HALO + blockIdx.x*blockDim.x*TILE_SIZE + i)*N 
                            + blockIdx.y*blockDim.y*TILE_SIZE + threadIdx.y;
            shSrc[i*smColDim + threadIdx.y] = src[idxLeftHalo];

            // Right halo
            int idxRightHalo = (HALO + blockIdx.x*blockDim.x*TILE_SIZE + i)*N 
                             + (blockIdx.y + 1)*blockDim.y*TILE_SIZE + HALO + threadIdx.y;
            shSrc[i*smColDim + (HALO + blockDim.y*TILE_SIZE + threadIdx.y)] = src[idxRightHalo];
        }
    }

    __syncthreads();

    // Applying stencil operations
#pragma unroll
    for (i = 0; i < TILE_SIZE; i++) {
        for (j = 0; j < TILE_SIZE; j++) {
            globalIndex = (HALO + blockIdx.x*blockDim.x*TILE_SIZE + threadIdx.x*TILE_SIZE + i)*N 
                        + blockIdx.y*blockDim.y*TILE_SIZE + threadIdx.y*TILE_SIZE + j + HALO;
            shMemIndex = (HALO + threadIdx.x*TILE_SIZE + i)*smColDim 
                       + HALO + threadIdx.y*TILE_SIZE + j;

            north = shSrc[shMemIndex - smColDim];
            south = shSrc[shMemIndex + smColDim];
            east  = shSrc[shMemIndex + 1];
            west  = shSrc[shMemIndex - 1];

            // Real stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }

    __syncthreads();
}