#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil37_hack2_cp_rows(double * dst, double * shared_rows, double *shared_cols, double *shared_slices, int d_xpitch, int d_ypitch, int d_zpitch, int s_xpitch, int s_ypitch, int s_zpitch, int n_rows, int n_cols, int n_slices, int tile_x, int tile_y, int tile_z) {

    int base_global_slice = tile_z * blockIdx.z;
    int base_global_row = tile_y * blockIdx.y;
    int base_global_col = blockDim.x * blockIdx.x;
    int dst_area = d_ypitch * d_xpitch;
    int s_area = gridDim.y * s_xpitch * 2;

    int base_global_idx = base_global_slice * dst_area + base_global_row * d_xpitch + base_global_col;

    int nextRow = base_global_row + 1;
    bool legalNextRow = nextRow < n_rows;

    int tx = threadIdx.x;
    bool legalCurCol = (base_global_col + tx) < n_cols;

    for (int tz = 0; tz < tile_z; ++tz) {
        bool legalCurSlice = (base_global_slice + tz) < n_slices;
        int idx_dst = base_global_idx + tz * dst_area + tx;
        int idx = (base_global_slice + tz) * s_area + blockIdx.y * s_xpitch * 2 + blockIdx.x * blockDim.x + tx;
        
        // Copy from dst to shared_rows if the column and slice are valid
        if (legalCurCol && legalCurSlice) {
            shared_rows[idx] = dst[idx_dst];
        }
        // Copy the next row if it is valid
        if (legalCurCol && legalCurSlice && legalNextRow) {
            shared_rows[idx + s_xpitch] = dst[idx_dst + d_xpitch];
        }
    }

    __syncthreads();
}