#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double *dst, double *src, int M, int N) {
    // Declare the shared memory array for source
    extern __shared__ double shared_mem[];
    double *shSrc = shared_mem;

    // Indexes
    int i, j;

    // Neighbor's values
    double north, south, east, west;

    // SharedMem Columns Dimension
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Thread indexes
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Block indexes
    int bx = blockIdx.x * blockDim.x;
    int by = blockIdx.y * blockDim.y;

    // Copying to shared memory: Inner part
    int globalIndex, shMemIndex;

    for (i = 0; i < TILE_SIZE; i++) {
        for (j = 0; j < TILE_SIZE; j++) {
            globalIndex = HALO * N + bx * TILE_SIZE * N + tx * TILE_SIZE * N + i * N + by * TILE_SIZE + ty * TILE_SIZE + j + HALO;
            shMemIndex = HALO * smColDim + tx * smColDim * TILE_SIZE + i * smColDim + HALO + ty * TILE_SIZE + j;
            shSrc[shMemIndex] = src[globalIndex];
        }
    }

    // Halos
    if (tx == 0 && ty == 0) {
        int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;

        // For Bottom and top row
        for (i = 0; i < HALO; i++) {
            for (j = 0; j < smColDim; j++) {
                indexTopHalo = (bx * TILE_SIZE + i) * N + by * TILE_SIZE + j;
                indexBottomHalo = (HALO + (bx + 1) * TILE_SIZE) * N + by * TILE_SIZE + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }

        // For right and left columns
        for (i = 0; i < HALO; i++) {
            for (j = 0; j < smRowDim - HALO * 2; j++) {
                indexLeftHalo = (HALO + bx * TILE_SIZE + j) * N + by * TILE_SIZE + i;
                indexRightHalo = (HALO + bx * TILE_SIZE + j) * N + ((by + 1) * TILE_SIZE) + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Stencil computation
    for (i = 0; i < TILE_SIZE; i++) {
        for (j = 0; j < TILE_SIZE; j++) {
            globalIndex = HALO * N + bx * TILE_SIZE * N + tx * TILE_SIZE * N + i * N + by * TILE_SIZE + ty * TILE_SIZE + j + HALO;
            shMemIndex = HALO * smColDim + tx * smColDim * TILE_SIZE + i * smColDim + HALO + ty * TILE_SIZE + j;

            // Getting the neighbors
            north = shSrc[shMemIndex - smColDim];
            south = shSrc[shMemIndex + smColDim];
            east  = shSrc[shMemIndex + 1];
            west  = shSrc[shMemIndex - 1];

            // Real Stencil operation
            dst[globalIndex] = (north + south + east + west) / 5.5;
        }
    }

    __syncthreads();
}