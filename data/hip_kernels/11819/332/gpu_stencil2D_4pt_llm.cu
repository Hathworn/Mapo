#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt(double *dst, double *src, int M, int N) {
    // Declare shared memory for source
    extern __shared__ double shared_mem[];
    double *shSrc = shared_mem;

    // Thread and Block index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Define shared memory dimensions
    int smColDim = HALO * 2 + blockDim.y * TILE_SIZE;
    int smRowDim = HALO * 2 + blockDim.x * TILE_SIZE;

    // Calculate global and shared memory indices for copying to shared memory
    int globalStartIndex = HALO * N + bx * blockDim.x * TILE_SIZE * N + tx * TILE_SIZE * N + by * blockDim.y * TILE_SIZE + ty * TILE_SIZE + HALO;
    int shMemStartIndex = HALO * smColDim + tx * smColDim * TILE_SIZE + HALO + ty * TILE_SIZE;

    // Copy to shared memory - Inner part
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            shSrc[shMemStartIndex + i * smColDim + j] = src[globalStartIndex + i * N + j];
        }
    }

    // Handle Halos
    if (tx == 0 && ty == 0) {
        int indexTopHalo, indexBottomHalo, indexLeftHalo, indexRightHalo;
        // Top and bottom rows
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smColDim; j++) {
                indexTopHalo = (bx * blockDim.x * TILE_SIZE + i) * N + by * blockDim.y * TILE_SIZE + j;
                indexBottomHalo = (HALO + (bx + 1) * blockDim.x * TILE_SIZE) * N + by * blockDim.y * TILE_SIZE + j;
                shSrc[i * smColDim + j] = src[indexTopHalo];
                shSrc[(HALO + blockDim.x * TILE_SIZE + i) * smColDim + j] = src[indexBottomHalo];
            }
        }
        // Left and right columns
        for (int i = 0; i < HALO; i++) {
            for (int j = 0; j < smRowDim - HALO * 2; j++) {
                indexLeftHalo = (HALO + bx * blockDim.x * TILE_SIZE + j) * N + by * blockDim.y * TILE_SIZE + i;
                indexRightHalo = (HALO + bx * blockDim.x * TILE_SIZE + j) * N + (by + 1) * blockDim.y * TILE_SIZE + HALO + i;
                shSrc[(HALO + j) * smColDim + i] = src[indexLeftHalo];
                shSrc[(HALO + j + 1) * smColDim - HALO + i] = src[indexRightHalo];
            }
        }
    }

    __syncthreads();

    // Perform stencil operation
    for (int i = 0; i < TILE_SIZE; i++) {
        for (int j = 0; j < TILE_SIZE; j++) {
            int shMemIndex = shMemStartIndex + i * smColDim + j;

            // Load neighbor values
            double north = shSrc[shMemIndex - smColDim];
            double south = shSrc[shMemIndex + smColDim];
            double east = shSrc[shMemIndex + 1];
            double west = shSrc[shMemIndex - 1];

            // Compute stencil result
            dst[globalStartIndex + i * N + j] = (north + south + east + west) / 5.5;
        }
    }
}