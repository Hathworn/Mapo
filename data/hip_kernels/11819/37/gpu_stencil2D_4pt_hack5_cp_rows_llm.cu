#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_stencil2D_4pt_hack5_cp_rows(double * dst, double * shared_cols, double * shared_rows, int tile_y, int M, int N) {
    #ifdef CUDA_DARTS_DEBUG
    if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0)) {
        printf("copy rows begin!\n");
    }
    #endif

    int base_global_row = tile_y * blockIdx.y;
    int base_global_col = blockDim.x * blockIdx.x;
    int base_global_idx = N * base_global_row + base_global_col;
    int t = threadIdx.x;

    bool legalCurCol = (base_global_col + t) < N;
    if (legalCurCol) {
        int idx = (base_global_row / tile_y) * 2 * N + t + base_global_col;
        // Coalesced memory access for aligned data
        shared_rows[idx] = dst[base_global_idx + t];

        int nextRow = base_global_row + 1;
        if (nextRow < M) {
            shared_rows[idx + N] = dst[base_global_idx + N + t];
        }
    }
    __syncthreads();

    #ifdef CUDA_CUDA_DEBUG
    if (blockIdx.y == 0 && blockIdx.x == 2 && (t == 0 || t == 1)) {
        printf("addr:%d, val = %f\n", (base_global_row / tile_y) * 2 * N + t + base_global_col + N, shared_rows[(base_global_row / tile_y) * 2 * N + t + base_global_col + N]);
    }
    #endif

    #ifdef CUDA_DARTS_DEBUG
    if ((blockIdx.x == 0) && (blockIdx.y == 0) && (threadIdx.x == 0)) {
        printf("copy rows finish!\n");
    }
    #endif
}