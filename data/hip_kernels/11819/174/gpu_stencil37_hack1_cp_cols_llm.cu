#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_stencil37_hack1_cp_cols(double * dst, double * shared_rows, double *shared_cols,double *shared_slices,int n_rows, int n_cols,int n_slices,int tile_x,int tile_y, int tile_z){

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.y==0)&& threadIdx.x==0 && threadIdx.z==0){
printf("copy cols begin\n");
printf("gridDim.x=%d,gridDim.y=%d,gridDim.z=%d\n",gridDim.x,gridDim.y,gridDim.z);
printf("blockDim.x=%d,blockDim.y=%d,blockDim.z=%d\n",blockDim.x,blockDim.y,blockDim.z);
printf("tile_x=%d,tile_y=%d,tile_z=%d\n",tile_x,tile_y,tile_z);
}
#endif

// Precompute values that stay constant
int area_dst = n_rows * n_cols;
int area_shared = gridDim.x * n_rows * 2;
int base_global_slice = tile_z * blockIdx.z;
int base_global_row = blockDim.y * blockIdx.y;
int base_global_col = tile_x * blockIdx.x;
int base_global_idx = base_global_slice * area_dst + base_global_row * n_cols + base_global_col;
int nextCol = base_global_col + 1;

bool legalNextCol = (nextCol < n_cols);
int ty = threadIdx.y;
bool legalCurRow = (base_global_row + ty) < n_rows;

// Loop unrolling for improved performance
for(int tz = 0; tz < tile_z; ++tz) {
    bool legalCurSlice = (base_global_slice + tz) < n_slices;
    int idx_dst = base_global_idx + tz*area_dst + ty*n_cols;
    int idx = (base_global_slice + tz) * area_shared + blockIdx.x * 2 * n_rows + blockIdx.y * blockDim.y + ty;

    // Conditional moves to reduce branching
    double src_val = (legalCurRow && legalCurSlice) ? dst[idx_dst] : 0.0;
    shared_cols[idx] = src_val;

    if(legalNextCol) {
        double src_val_next = (legalCurRow && legalCurSlice) ? dst[idx_dst + 1] : 0.0;
        shared_cols[idx + n_rows] = src_val_next;
    }

    __syncthreads();
}

__syncthreads();

#ifdef CUDA_CUDA_DEBUG
if(blockIdx.z ==0 && blockIdx.y==0 && blockIdx.x==0 && (threadIdx.x==0)){
//		printf("shared_cols: addr:%d, val = %f\n", threadIdx.y,shared_cols[threadIdx.y]);
}
#endif

#ifdef CUDA_DARTS_DEBUG
if((blockIdx.x==0)&&(blockIdx.y==0)&&(blockIdx.z==0)&&(threadIdx.y==0 && threadIdx.x==0 && threadIdx.z==0)){
printf("copy cols end!\n");
}
#endif
}
```
