#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Precompute variables
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    // Shift output and input pointers to the current plane
    output += o * output_w * output_h;
    input += i * input_w * input_h;

    // Load weight and bias
    float the_weight = weight[k];
    float the_bias = bias[k];

    // Calculate pixel positions
    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y * gridDim.y;

    // Process each output pixel
    for (int yy = yy_start; yy < yy_end; yy += yy_step)
    {
        for (int xx = xx_start; xx < xx_end; xx += xx_step)
        {
            // Initialize sum for averaging
            float sum = 0.0f;
            int input_base_idx = yy * dH * input_w + xx * dW;

            // Accumulate values over kernel window
            for (int ky = 0; ky < kH; ky++)
            {
                for (int kx = 0; kx < kW; kx++)
                {
                    sum += input[input_base_idx + ky * input_w + kx]; // Accumulate input values
                }
            }

            // Store result in output
            output[yy * output_w + xx] = the_weight * sum + the_bias;
        }
    }
}