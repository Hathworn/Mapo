#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradinput(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate the start index and step for each thread in the block
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int yy_start = blockIdx.y * blockDim.y + threadIdx.y;
  
    // Store weight value
    float the_weight = weight[k];
    
    // Offset the pointers for the current block 
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    
    // Use shared memory for storing partial calculations
    __shared__ float shared_gradOutput[8192];  // example size, adjust according to kernel
    __shared__ float shared_gradInput[8192];   // example size, adjust according to kernel
    
    // Load data into shared memory
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    shared_gradOutput[tid] = gradOutput[yy_start * output_w + xx_start];
    __syncthreads();
    
    // Perform calculations
    float z = shared_gradOutput[tid] * the_weight;
    for (int ky = 0; ky < kH; ky++) {
        for (int kx = 0; kx < kW; kx++) {
            // Use atomic add to prevent race conditions
            atomicAdd(&shared_gradInput[yy_start * dH * input_w + xx_start * dW + kx + ky * input_w], z);
        }
    }
    __syncthreads();
    
    // Copy results from shared memory back to global memory
    gradInput[yy_start * dH * input_w + xx_start * dW] = shared_gradInput[tid];
}