#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Compute output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;

    // Get weight
    float the_weight = weight[k];

    // Optimize: Use shared memory to reduce global memory access overhead
    __shared__ float shared_gradOutput[1024]; // assuming blockDim.x * blockDim.y <= 1024
    int thread_pos = threadIdx.y * blockDim.x + threadIdx.x;
    if (thread_pos < output_w * output_h) {
        shared_gradOutput[thread_pos] = gradOutput[thread_pos] * the_weight;
    }
    __syncthreads();

    // Compute gradInput
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            float z = shared_gradOutput[yy * output_w + xx];
            int base_index = yy * dH * input_w + xx * dW;
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    atomicAdd(&(gradInput[base_index + ky * input_w + kx]), z);
                }
            }
        }
    }
}