#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // Compute offsets based on thread/block ID
    int o = blockIdx.x;

    // Calculate thread positioning
    int xx = threadIdx.x;
    int yy = blockIdx.y * blockDim.y + threadIdx.y;

    // Select input/output plane
    float *gradOutput_plane = gradOutput + o * output_w * output_h;
    float *gradInput_plane = gradInput + o * input_w * input_h;
    float *indices_x_plane = indices_x + o * output_w * output_h;
    float *indices_y_plane = indices_y + o * output_w * output_h;

    // Stride over the y dimension
    for (int yy_stride = yy; yy_stride < output_h; yy_stride += blockDim.y * gridDim.y) {
        int y_start = (int)floorf((float)yy_stride / output_h * input_h);

        // Stride over the x dimension
        for (int xx_stride = xx; xx_stride < output_w; xx_stride += blockDim.x) {
            int x_start = (int)floorf((float)xx_stride / output_w * input_w);

            float z = gradOutput_plane[yy_stride * output_w + xx_stride];

            int argmax_x = (int)indices_x_plane[yy_stride * output_w + xx_stride] - 1;
            int argmax_y = (int)indices_y_plane[yy_stride * output_w + xx_stride] - 1;

            atomicAdd(&(gradInput_plane[argmax_x + argmax_y * input_w]), z); // Use atomicAdd for correct concurrency handling
        }
    }
}