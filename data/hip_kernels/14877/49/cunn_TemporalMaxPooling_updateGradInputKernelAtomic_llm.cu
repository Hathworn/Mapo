#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Pre-calculate offsets for each input pointer to improve readability
    int batch_idx = blockIdx.x;
    int time_idx = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    
    // Only proceed with relevant threads
    if (time_idx < output_w) {
        float *gradInput_data = gradInput + batch_idx * input_w * input_n + time_idx * input_n * dW;
        float *gradOutput_data = gradOutput + batch_idx * output_w * input_n + time_idx * input_n;
        float *indices_data = indices + batch_idx * output_w * input_n + time_idx * input_n;

        // Utilize local memory for indices within this thread context
        for (int feat = 0; feat < input_n; ++feat) {
            int index = static_cast<int>(indices_data[feat]);
            atomicAdd(&gradInput_data[index * input_n + feat], gradOutput_data[feat]);
        }
    }
}