#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_LookupTable_accGradParametersKernel(float *input, float *indices, float *gradOutput, float *gradWeight, float *count, float defaultScale, long numel, long stride, int paddingValue) {

    // Improved block size calculation for better load balancing
    int idx = blockIdx.x * blockDim.y + threadIdx.y;

    // Exit early if bounds are exceeded
    if (idx >= numel || input[idx] == paddingValue) return;

    // Process only new input values (not already processed by a previous warp)
    if (idx == 0 || input[idx] != input[idx - 1]) {
        
        // Constants are precalculated by the compiler, avoiding recalculations
        const int SZ = 4;
        const int startFeature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
        const int weightRow = ((int) input[idx] - 1) * stride;
        const int gradOutputRow = ((int) indices[idx] - 1) * stride;
        const float scale = count ? defaultScale / count[idx] : defaultScale;

        // Using shared memory for caching (thread local in global scope to avoid bank conflicts)
        float gradient[SZ];
        float weight[SZ];

        #pragma unroll
        for (int ii = 0; ii < SZ; ii++) {
            int featureDim = startFeature + ii * WARP_SIZE;
            if (featureDim < stride) {
                gradient[ii] = gradOutput[gradOutputRow + featureDim];
                weight[ii] = gradWeight[weightRow + featureDim];
            }
        }

        #pragma unroll
        for (int ii = 0; ii < SZ; ii++) {
            weight[ii] += gradient[ii] * scale;
        }

        #pragma unroll
        for (int ii = 0; ii < SZ; ii++) {
            int featureDim = startFeature + ii * WARP_SIZE;
            if (featureDim < stride) {
                gradWeight[weightRow + featureDim] = weight[ii];
            }
        }

        // Process subsequent inputs with the same value
        while (++idx < numel && input[idx] == input[idx - 1]);
    }
}