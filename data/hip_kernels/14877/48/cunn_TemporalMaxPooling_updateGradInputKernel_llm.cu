#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate the starting index for thread in a more readable manner
    int threadIndex = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (threadIndex < output_w) {
        int batchOffset = blockIdx.x * input_w * input_n;
        int outputOffset = blockIdx.x * output_w * input_n + threadIndex * input_n;

        float *gradInput_data = gradInput + batchOffset + threadIndex * kW * input_n;
        float *gradOutput_data = gradOutput + outputOffset;
        float *indices_data = indices + outputOffset;

        // Unrolling the loop for potential performance gain
        #pragma unroll
        for (int feat = 0; feat < input_n; ++feat) {
            int index = static_cast<int>(indices_data[feat]);
            atomicAdd(&gradInput_data[index * input_n + feat], gradOutput_data[feat]);
        }
    }
}