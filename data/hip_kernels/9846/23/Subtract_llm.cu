#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Calculate x and y using optimized formula 
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate linear index
    int p = y * width + x;

    // Check bounds and perform subtraction
    if (x < width && y < height)
        d_Result[p] = d_Data1[p] - d_Data2[p];
}