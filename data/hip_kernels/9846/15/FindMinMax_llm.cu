#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * blockDim.x + tx;
    const int y = blockIdx.y * blockDim.y;
    const int b = blockDim.x;
    int p = y * pitch + x;

    // Initialize shared memory with data values conditionally
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        minvals[tx] = INFINITY;  // Set to infinity for comparison
        maxvals[tx] = -INFINITY; // Set to negative infinity for comparison
    }

    // Loop unroll optimization
    #pragma unroll
    for (int ty = 1; ty < 16; ++ty) {
        p += pitch;
        if (x < width) {
            float val = d_Data[p];
            if (val < minvals[tx]) minvals[tx] = val;
            if (val > maxvals[tx]) maxvals[tx] = val;
        }
    }
    __syncthreads();

    // Shared memory reduction with loop unrolling
    int mod = 1;
    for (int d = 1; d < b; d <<= 1) {
        if ((tx & mod) == 0) {
            if (minvals[tx + d] < minvals[tx])
                minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx])
                maxvals[tx] = maxvals[tx + d];
        }
        mod = mod * 2 + 1;
        __syncthreads();
    }

    // Output final min and max values
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}