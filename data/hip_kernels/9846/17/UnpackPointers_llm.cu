#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    // Calculate global indices
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    const int by = blockIdx.y;
    const int blockWidth = 16; // Original x increment value

    int numPts = 0;

    // Adjust loop bounds for blocks
    for (int y = by * (h / (gridDim.y)); y < (by + 1) * (h / (gridDim.y)); y++) {
        for (int x = bx * blockWidth; x < (bx + 1) * blockWidth && x < w; x += 16) {
            unsigned int val = minmax[y * w + x + tx];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if ((val & 0x1) && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}