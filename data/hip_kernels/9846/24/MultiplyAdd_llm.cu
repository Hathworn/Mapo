#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate global thread index using efficient built-in methods
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;

    // Ensure thread operates within bounds
    if (x < width && y < height) {
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
}