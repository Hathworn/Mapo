#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOrientations(float *g_Data, float *d_Sift, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize histogram
    for (int i = 0; i < 13; i++)
        hist[i * 32 + tx] = 0.0f;
    __syncthreads();

    // Precompute Gaussian values
    if (tx < 15) {
        float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));
    }

    int xp = (int)(d_Sift[bx + 0 * maxPts] - 6.5f);
    int yp = (int)(d_Sift[bx + 1 * maxPts] - 6.5f);
    int px = xp & 15;  // Improved modulus operation by bitwise mask
    int x = tx - px;

    // Load data into shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = max(0, min(w - 1, xp + x));
        int yi = max(0, min(h - 1, yp + y));
        if (x >= 0 && x < 15) 
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Calculate gradients and build histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = 16.0f * atan2f(dy, dx) / 3.1416f + 16.5f;
            if (bin == 32) bin = 0;
            float grad = sqrtf(dx * dx + dy * dy);
            atomicAdd(&hist[32 * (x - 1) + bin], grad * gauss[x] * gauss[y]);  // Use atomicAdd to avoid race condition
        }
    }
    __syncthreads();

    // Histogram folding
    for (int shift = 8; shift > 0; shift >>= 1) {
        if (tx < shift * 4) 
            hist[tx] += hist[tx + shift * 32];
        __syncthreads();
    }
    if (tx == 0) {
        hist[32] = 6 * hist[0] + 4 * (hist[1] + hist[31]) + (hist[2] + hist[30]);
        hist[63] = 6 * hist[31] + 4 * (hist[0] + hist[30]) + (hist[1] + hist[29]);
    }
    if (tx >= 1 && tx <= 30) 
        hist[tx + 32] = 6 * hist[tx] + 4 * (hist[tx + 1] + hist[tx - 1]) + 
                        (hist[tx + 2] + hist[tx - 2]);

    __syncthreads();

    // Non-max suppression
    float v = hist[32 + tx];
    hist[tx] = (v > hist[32 + ((tx + 1) & 31)] && v >= hist[32 + ((tx + 31) & 31)]) ? v : 0.0f;
    __syncthreads();

    // Find the key peaks
    if (tx == 0) {
        float maxval1 = 0.0f;
        float maxval2 = 0.0f;
        int i1 = -1, i2 = -1;
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }
        float val1 = hist[32 + ((i1 + 1) & 31)];
        float val2 = hist[32 + ((i1 + 31) & 31)];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Sift[bx + 5 * maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);

        if (maxval2 < 0.8f * maxval1) { i2 = -1; }
        if (i2 >= 0) {
            val1 = hist[32 + ((i2 + 1) & 31)];
            val2 = hist[32 + ((i2 + 31) & 31)];
            float peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Sift[bx + 6 * maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        } else
            d_Sift[bx + 6 * maxPts] = i2;
    }
}