#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    // Calculate global thread position
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Utilize shared memory (if necessary, based on additional requirements or data reuse)
    // Declare shared memory if needed: __shared__ float sharedData[];

    // Check bounds and perform subtraction
    if (x < width && y < height) {
        int p = y * pitch + x;
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    
    // Remove unnecessary synchronization if no shared memory use
    // __syncthreads(); // Removed as no shared memory operations require synchronization
}