#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Calculate global x and y position
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate slice size and position
    int sz = height * pitch;
    int p = threadIdx.z * sz + y * pitch + x;

    // Ensure x and y are within bounds before performing computation
    if (x < width && y < height) {
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
}