#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int pitch, int height) {
    // Shared memory for data cache
    __shared__ float data1[3 * (MINMAX_W + 2)];
    __shared__ float data2[3 * (MINMAX_W + 2)];
    __shared__ float data3[3 * (MINMAX_W + 2)];
    __shared__ float ymin1[MINMAX_W + 2];
    __shared__ float ymin2[MINMAX_W + 2];
    __shared__ float ymin3[MINMAX_W + 2];
    __shared__ float ymax1[MINMAX_W + 2];
    __shared__ float ymax2[MINMAX_W + 2];
    __shared__ float ymax3[MINMAX_W + 2];

    // Pre-calculate indices and limits
    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_W;
    const int xEnd = xStart + MINMAX_W - 1;
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;
    const int xEndClamped = min(xEnd, width - 1);
    const int memWid = MINMAX_W + 2;

    int memPos0 = tx - WARP_SIZE + 1;
    int memPos1 = tx - WARP_SIZE + 1;
    int yq = 0;
    unsigned int output = 0;

    for (int y = 0; y < 32 + 2; y++) {
        output >>= 1;
        int memPos = yq * memWid + (tx - WARP_SIZE + 1);
        int yp = 32 * blockIdx.y + y - 1;
        yp = max(yp, 0);
        yp = min(yp, height - 1);
        int readStart = yp * pitch;

        // Load data into shared memory
        if (tx >= (WARP_SIZE - 1)) {
            if (xReadPos < 0 || xReadPos >= width) {
                data1[memPos] = data2[memPos] = data3[memPos] = 0;
            } else {
                data1[memPos] = d_Data1[readStart + xReadPos];
                data2[memPos] = d_Data2[readStart + xReadPos];
                data3[memPos] = d_Data3[readStart + xReadPos];
            }
        }
        __syncthreads();

        // Calculate local min and max
        int memPos2 = yq * memWid + tx;
        if (y > 1 && tx < memWid) {
            float min1 = fminf(fminf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float min2 = fminf(fminf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float min3 = fminf(fminf(data3[memPos0], data3[memPos1]), data3[memPos2]);
            float max1 = fmaxf(fmaxf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float max2 = fmaxf(fmaxf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float max3 = fmaxf(fmaxf(data3[memPos0], data3[memPos1]), data3[memPos2]);
            ymin2[tx] = fminf(fminf(min1, min2), min3);
            ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
        }
        __syncthreads();

        // Perform comparison and update output
        if (y > 1 && tx < MINMAX_W && xWritePos <= xEndClamped) {
            float minv = fminf(fminf(fminf(fminf(fminf(ymin2[tx], ymin2[tx + 2]), ymin1[tx + 1]), ymin3[tx + 1]), data2[memPos0 + 1]), data2[memPos2 + 1]);
            minv = fminf(minv, d_Threshold[1]);
            float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(ymax2[tx], ymax2[tx + 2]), ymax1[tx + 1]), ymax3[tx + 1]), data2[memPos0 + 1]), data2[memPos2 + 1]);
            maxv = fmaxf(maxv, d_Threshold[0]);
            if (data2[memPos1 + 1] < minv || data2[memPos1 + 1] > maxv) {
                output |= 0x80000000;
            }
        }
        __syncthreads();

        // Rotate shared memory index
        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }

    // Write output results
    if (tx < MINMAX_W && xWritePos < width) {
        int writeStart = blockIdx.y * pitch + xWritePos;
        d_Result[writeStart] = output;
    }
}