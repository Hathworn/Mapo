#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    // Using shared memory to reduce global memory access latency
    __shared__ unsigned int sharedVal[32];
    
    const int tx = threadIdx.x;
    int numPts = 0;

    for (int y = 0; y < h / 32; y++) {
        for (int x = 0; x < w; x += 16) {
            sharedVal[tx] = minmax[y * w + x + tx];
            __syncthreads(); // Ensure all threads have loaded to sharedVal

            unsigned int val = sharedVal[tx];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    // Check bit and the numPts condition in a single evaluation
                    if ((val & 0x1) && (numPts < maxPts)) {
                        // Avoid bank conflicts when writing to ptrs
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
            
            __syncthreads(); // Ensure all threads have finished processing the segment
        }
    }
}