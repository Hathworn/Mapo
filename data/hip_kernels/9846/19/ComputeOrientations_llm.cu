#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize histogram to zero
    for (int i = tx; i < 13 * 32; i += blockDim.x)
        hist[i] = 0.0f;
    __syncthreads();

    // Calculate Gaussian kernel values for smoothing
    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15)
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));
    __syncthreads();

    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;
    int px = xp & 15;
    int x = tx - px;

    // Load data into shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = xp + x;
        int yi = yp + y;
        xi = max(0, min(w - 1, xi));
        yi = max(0, min(h - 1, yi));
        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Calculate gradients and update histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = __float2int_rd(16.0f * atan2f(dy, dx) / 3.1416f + 16.5f) % 32;
            float grad = sqrtf(dx*dx + dy*dy);
            atomicAdd(&hist[32 * (x - 1) + bin], grad * gauss[x] * gauss[y]);
        }
    }
    __syncthreads();

    // Reduce histogram values along y-axis
    for (int y = 0; y < 6; y++) {
        if (tx < 32) {
            hist[y * 32 + tx] += hist[(y + 8) * 32 + tx];
            if (y < 4) hist[y * 32 + tx] += hist[(y + 4) * 32 + tx];
            if (y < 2) hist[y * 32 + tx] += hist[(y + 2) * 32 + tx];
            hist[tx] += hist[32 + tx];
        }
    }
    __syncthreads();

    // Smooth histogram using weighted 1D convolution
    float v = hist[32 + tx];
    hist[tx] = (v > hist[32 + ((tx + 1) & 31)] && v >= hist[32 + ((tx + 31) & 31)] ? v : 0.0f);
    __syncthreads();

    // Find peaks in the histogram
    if (tx == 0) {
        float maxval1 = 0.0f;
        float maxval2 = 0.0f;
        int i1 = -1, i2 = -1;
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1; maxval1 = v; i2 = i1; i1 = i;
            } else if (v > maxval2) {
                maxval2 = v; i2 = i;
            }
        }

        // Compute subpixel orientation for first peak
        float val1 = hist[32 + ((i1 + 1) & 31)];
        float val2 = hist[32 + ((i1 + 31) & 31)];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Orient[bx] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);

        // Compute subpixel orientation for second peak, if valid
        if (maxval2 < 0.8f * maxval1)
            i2 = -1;
        if (i2 >= 0) {
            val1 = hist[32 + ((i2 + 1) & 31)];
            val2 = hist[32 + ((i2 + 31) & 31)];
            peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Orient[bx + maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        } else {
            d_Orient[bx + maxPts] = -1;
        }
    }
}