#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;  // Simplified arithmetic
    const int y = blockIdx.y * 16;
    const int b = blockDim.x;
    int p = y * width + x;
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        minvals[tx] = FLT_MAX;  // Use FLT_MAX and FLT_MIN to avoid unnecessary read
        maxvals[tx] = -FLT_MAX;
    }

    for (int ty = 1; ty < 16; ty++) {
        p += width;
        if (x < width) {
            float val = d_Data[p];
            if (val < minvals[tx])
                minvals[tx] = val;
            if (val > maxvals[tx])
                maxvals[tx] = val;
        }
    }

    __syncthreads();
    for (int d = b / 2; d > 0; d >>= 1) {  // Use a single loop for reduction
        if (tx < d) {
            if (minvals[tx + d] < minvals[tx])
                minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx])
                maxvals[tx] = maxvals[tx + d];
        }
        __syncthreads();
    }

    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}