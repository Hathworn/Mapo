#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassColMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVCOL_W*(CONVCOL_H + 2*RADIUS)];
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int block = blockIdx.x/(NUM_SCALES+3);
    const int scale = blockIdx.x - (NUM_SCALES+3)*block;
    const int miny = blockIdx.y*CONVCOL_H;
    const int maxy = min(miny + CONVCOL_H, height) - 1;
    const int totStart = miny - RADIUS;
    const int totEnd = maxy + RADIUS;
    const int colStart = block*CONVCOL_W + tx;
    const int colEnd = colStart + (height-1)*pitch;
    const int sStep = CONVCOL_W*CONVCOL_S;
    const int gStep = pitch*CONVCOL_S;
    float *kernel = d_Kernel + scale*16;
    const int size = pitch*height*scale;
    d_Result += size;
    d_Data += size;

    if (colStart < width) {
        float *sdata = data + ty*CONVCOL_W + tx;
        int gPos = colStart + (totStart + ty)*pitch;
        for (int y = totStart + ty; y <= totEnd; y += blockDim.y) {
            if (y < 0)
                sdata[0] = d_Data[colStart];
            else if (y >= height)
                sdata[0] = d_Data[colEnd];
            else
                sdata[0] = d_Data[gPos];
            sdata += sStep;
            gPos += gStep;
        }
    }

    __syncthreads();

    if (colStart < width) {
        float *sdata = data + ty*CONVCOL_W + tx;
        int gPos = colStart + (miny + ty)*pitch;
        for (int y = miny + ty; y <= maxy; y += blockDim.y) {
            // Use local variables to minimize array access and improve performance
            float val0 = sdata[0*CONVCOL_W];
            float val1 = sdata[1*CONVCOL_W];
            float val2 = sdata[2*CONVCOL_W];
            float val3 = sdata[3*CONVCOL_W];
            float val4 = sdata[4*CONVCOL_W];
            float val5 = sdata[5*CONVCOL_W];
            float val6 = sdata[6*CONVCOL_W];
            float val7 = sdata[7*CONVCOL_W];
            float val8 = sdata[8*CONVCOL_W];

            d_Result[gPos] = 
                (val0 + val8) * kernel[0] + 
                (val1 + val7) * kernel[1] + 
                (val2 + val6) * kernel[2] + 
                (val3 + val5) * kernel[3] + 
                val4 * kernel[4];
            sdata += sStep;
            gPos += gStep;
        }
    }
}