#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UpdateSurface(hipSurfaceObject_t surf, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (y >= height || x >= width) return;  // Correct condition operator to logical OR

    // Precompute commonly used values outside of branches
    float cosTime = __cosf(time) * 0.5f;
    float xVar = static_cast<float>(x) / static_cast<float>(width);
    float yVar = static_cast<float>(y) / static_cast<float>(height);
    float costx = cosTime + xVar;
    float costyMany = __cosf(static_cast<float>(x) / 100.0f * time) * 0.5f + xVar;
    
    float4 pixel;

    // Use flattening to reduce if-else depth for performance
    if (y == 0) {
        pixel = make_float4(costyMany * 0.3f, costyMany, costyMany * 0.4f, 1.0f);
    } else if (y == height - 1) {
        pixel = make_float4(costyMany * 0.6f, costyMany * 0.7f, costyMany, 1.0f);
    } else if (x % 2 == 0) {
        pixel = (x > width / 2) ?
                  make_float4(0.1f, 0.5f, costx, 1.0f) :
                  make_float4(costx, 0.1f, 0.2f, 1.0f);
    } else if (x > width - 2 || x <= 1) {
        float costxMany = __cosf(y * time) * 0.5f + yVar;
        pixel = make_float4(costxMany, costxMany * 0.9f, costxMany * 0.6f, 1.0f);
    } else {
        pixel = make_float4(costx * 0.3f, costx * 0.4f, costx * 0.6f, 1.0f);
    }

    surf2Dwrite(pixel, surf, x * sizeof(float4), y);  // Ensure the correct size to prevent misalignment
}