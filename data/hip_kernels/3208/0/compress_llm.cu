#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 9
#define K N/3
#define ThreadsPerBlock K
#define NumBlocks K

__global__ void compress(float *mat, int n, float *comp, int k) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    // Only proceed if within matrix bounds
    if (row < k && col < k) {
        float sum = 0.0f; // Initialize sum for each element
        for (int i_row = 0; i_row < k; i_row++) {
            for (int j_col = 0; j_col < k; j_col++) {
                sum += mat[(col + j_col) + (row + i_row) * n]; // Accumulate values
            }
        }
        comp[col + row * k] = sum; // Assign accumulated sum to output
    }
}