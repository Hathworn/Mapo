#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpuSum(int *a, int *b, int *c, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Optimize memory access by using shared memory
    __shared__ int shared_a[256];
    __shared__ int shared_b[256];

    while (idx < n) {
        // Load data into shared memory
        shared_a[threadIdx.x] = a[idx];
        shared_b[threadIdx.x] = b[idx];
        __syncthreads();

        // Perform computation using shared memory
        c[idx] = shared_a[threadIdx.x] + shared_b[threadIdx.x];
        __syncthreads();

        idx += blockDim.x * gridDim.x;
    }
}