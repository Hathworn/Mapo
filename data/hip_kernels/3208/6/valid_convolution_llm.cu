#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void valid_convolution(float *d_kernel, int k_size, float *d_matrix, int size_x, int size_y, float *d_conv, int max_row, int max_col) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    // Check if thread is within bounds
    if (max_row > row && max_col > col) {
        // Initialize the convolution sum to 0
        float sum = 0.0f;

        // Perform convolution
        for (int k_row = 0; k_row < k_size; k_row++) {
            for (int k_col = 0; k_col < k_size; k_col++) {
                int matrix_idx = (col + k_col) + (row + k_row) * size_x;
                int kernel_idx = k_col + k_row * k_size;
                sum += d_kernel[kernel_idx] * d_matrix[matrix_idx];
            }
        }

        // Write result to global memory
        d_conv[col + row * max_col] = sum;
    }
}