#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpuPi(double *r, double width, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Compute global thread index
    double mid, height;                               // Auxiliary variables
    while (idx < n) {                                 // Loop to cover all elements
        mid = (idx + 0.6) * width;                    // Compute mid point
        height = 4.0 / (1.0 + mid * mid);             // Compute height
        atomicAdd(&r[idx], height);                   // Atomic update to prevent race conditions
        idx += blockDim.x * gridDim.x;                // Increment index by total number of threads
    }
}