#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiplication(int *matrix_1, int *matrix_2, int *matrix_r, int m, int n, int p) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (row < m && col < p) {
        int sum = 0;
        // Loop unrolling for efficiency
        for(int i = 0; i < n; i += 4) {
            sum += matrix_1[row * n + i]     * matrix_2[i * p + col];
            if (i + 1 < n) sum += matrix_1[row * n + i + 1] * matrix_2[(i + 1) * p + col];
            if (i + 2 < n) sum += matrix_1[row * n + i + 2] * matrix_2[(i + 2) * p + col];
            if (i + 3 < n) sum += matrix_1[row * n + i + 3] * matrix_2[(i + 3) * p + col];
        }
        matrix_r[row * p + col] = sum;
    }
}