#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReducePI2(float* d_sum, int num, float* d_pi) {
    int id = threadIdx.x;
    extern __shared__ float s_sum[];

    // Load data into shared memory and synchronize
    s_sum[id] = d_sum[id];
    __syncthreads();

    // Use warp-synchronous method for reduction
    for (int i = blockDim.x >> 1; i > 0; i >>= 1) {
        if (id < i) {
            s_sum[id] += s_sum[id + i];
        }
        __syncthreads();
    }

    // Output final reduced sum and calculate pi
    if (id == 0) {
        *d_pi = s_sum[0] / num;
        printf("%d,%f\n", id, *d_pi);
    }
}