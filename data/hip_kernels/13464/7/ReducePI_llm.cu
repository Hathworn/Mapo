#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReducePI(float* d_sum, int num) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int gid = id;
    float temp;
    extern __shared__ float s_pi[];

    // Initialize shared memory for partial sums
    s_pi[threadIdx.x] = 0.f;

    // Accumulate partial sums in shared memory
    while (gid < num) {
        temp = (gid + 0.5f) / num;
        s_pi[threadIdx.x] += 4.0f / (1 + temp * temp);
        gid += blockDim.x * gridDim.x; // Fix the increment to ensure correct iteration
    }
    __syncthreads(); // Synchronize before reduction

    // Perform reduction within the block using shared memory
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            s_pi[threadIdx.x] += s_pi[threadIdx.x + i];
        }
        __syncthreads(); // Synchronize threads after each step of reduction
    }

    // Write block's sum to the output
    if (threadIdx.x == 0) {
        d_sum[blockIdx.x] = s_pi[0];
    }
}