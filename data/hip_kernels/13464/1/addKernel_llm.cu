#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate unique thread index for global memory access
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform addition for each thread
    c[i] = a[i] + b[i];
}