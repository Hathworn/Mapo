#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < blockDim.x) // Ensure within bounds
    {
        c[i] = a[i] + b[i]; // Perform addition
    }
}