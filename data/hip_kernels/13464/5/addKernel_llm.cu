#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; // Extend index for block/grid
    int stride = blockDim.x * gridDim.x; // Stride for grid

    // Unroll loop for better performance by using stride
    for (int idx = i; idx < 1024 * 500; idx += stride) {
        c[idx] = a[idx] * 10 + b[idx] * 5;
    }
}