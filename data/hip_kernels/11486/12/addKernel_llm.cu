#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within array bounds (assuming max array size is known)
    if (i < SIZE) {
        c[i] = a[i] + b[i];
    }
}