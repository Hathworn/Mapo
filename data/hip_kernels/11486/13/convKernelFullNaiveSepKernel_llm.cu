#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convKernelFullNaiveSepKernel(float* d_Input, float* d_Output, float* d_Kernel, int imageW, int imageH, int kernelR)
{
    // Compute row and column index
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    // Compute global index location
    int loc = row * imageW + col;

    // Check if within image bounds
    if (row >= imageH || col >= imageW) return;

    float s = 0.0f;
    // Iterate over the kernel
    for (int i = -kernelR; i <= kernelR; i++) {
        for (int j = -kernelR; j <= kernelR; j++) {
            int currentRow = row + i;
            int currentCol = col + j;
            // Accumulate if indices within bounds
            if (currentRow >= 0 && currentRow < imageH && currentCol >= 0 && currentCol < imageW) {
                float inputVal = d_Input[currentRow * imageW + currentCol];
                s += inputVal * d_Kernel[kernelR - i] * d_Kernel[kernelR - j];
            }
        }
    }
    // Store the result in the output
    d_Output[loc] = s;
}