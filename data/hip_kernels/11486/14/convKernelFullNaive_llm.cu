#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convKernelFullNaive(float* d_Input, float* d_Output, float* d_Kernel, int imageW, int imageH, int kernelR)
{
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    int loc = row * imageW + col;

    float s = 0;
    
    // Cache kernel radius multiplication factor
    int kernelSize = kernelR * 2 + 1;

    for (int i = -kernelR; i <= kernelR; i++) {
        for (int j = -kernelR; j <= kernelR; j++) {
            int newRow = row + i;
            int newCol = col + j;

            // Check bounds and perform computation
            if (newRow >= 0 && newRow < imageH && newCol >= 0 && newCol < imageW) {
                int inputIdx = newRow * imageW + newCol;
                int kernelIdx = (kernelR - i) * kernelSize + (kernelR - j);
                s += d_Input[inputIdx] * d_Kernel[kernelIdx];
            }
        }
    }
    d_Output[loc] = s;
}