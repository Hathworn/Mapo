#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Using block and thread index to potentially handle larger arrays
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}