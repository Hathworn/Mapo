#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertPitchedFloatToRGBA_kernel(uchar4 *out_image, const float *in_image, int width, int height, int pitch, float lowerLim, float upperLim) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified block and thread index calculation
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    uchar4 temp;
    if (x < width && y < height) {
        float val = *((float *)((char *)in_image + y * pitch) + x);

        // Use white color for unmatched pixels
        if (!isfinite(val)) {
            temp.x = 255;
            temp.y = 255;
            temp.z = 255;
            temp.w = 255;
        } else {
            // Rescale value from [lowerLim, upperLim] to [0, 1]
            val = (val - lowerLim) / (upperLim - lowerLim);

            float r = 1.0f;
            float g = 1.0f;
            float b = 1.0f;
            if (val < 0.25f) {
                r = 0;
                g = 4.0f * val;
            } else if (val < 0.5f) {
                r = 0;
                b = 1.0f + 4.0f * (0.25f - val);
            } else if (val < 0.75f) {
                r = 4.0f * (val - 0.5f);
                b = 0;
            } else {
                g = 1.0f + 4.0f * (0.75f - val);
                b = 0;
            }
            temp.x = static_cast<unsigned char>(255.0f * r);  // Cast to uchar for clarity
            temp.y = static_cast<unsigned char>(255.0f * g);
            temp.z = static_cast<unsigned char>(255.0f * b);
            temp.w = 255;
        }
        out_image[y * width + x] = temp;  // Improved out_image indexing
    }
}