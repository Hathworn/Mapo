#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deInterleave_kernel2(float *d_X_out, float *d_Y_out, char *d_XY_in, int pitch_out, int pitch_in, int width, int height) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) { // Check within image bounds
        char *input_ptr = d_XY_in + y * pitch_in; // Precompute address
        float *data = (float *)(input_ptr) + 2 * x;
        
        char *output_ptr_x = (char *)d_X_out + y * pitch_out; // Precompute output address for X
        *((float *)output_ptr_x + x) = data[0];

        char *output_ptr_y = (char *)d_Y_out + y * pitch_out; // Precompute output address for Y
        *((float *)output_ptr_y + x) = data[1];
    }
}