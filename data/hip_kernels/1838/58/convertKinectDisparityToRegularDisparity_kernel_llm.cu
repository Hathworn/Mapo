#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertKinectDisparityToRegularDisparity_kernel(float *d_regularDisparity, int d_regularDisparityPitch, const float *d_KinectDisparity, int d_KinectDisparityPitch, int width, int height) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread is within the image bounds
    if ((x < width) && (y < height)) {
        // Calculate disparity inputs and outputs
        float d_in = *reinterpret_cast<const float*>(
            reinterpret_cast<const char*>(d_KinectDisparity) + y * d_KinectDisparityPitch + x * sizeof(float));

        float d_out = (d_in == 0.0f) ? nanf("") : -d_in;

        *reinterpret_cast<float*>(
            reinterpret_cast<char*>(d_regularDisparity) + y * d_regularDisparityPitch + x * sizeof(float)) = d_out;
    }
}
```
