#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normal_eqs_disparity_GPU(float *d_CD, const float *d_disparity_compact, const float4 *d_Zbuffer_normals_compact, const int *d_ind_disparity_Zbuffer, float fx, float fy, float ox, float oy, float b, int n_cols, const int *d_n_values_disparity, const int *d_start_ind_disparity, float w_disp) {

    int n_val_accum = gridDim.x * blockDim.x;
    int n_disparity = d_n_values_disparity[blockIdx.y];
    int n_accum = (int)ceilf((float)n_disparity / (float)n_val_accum);
    int start_ind = d_start_ind_disparity[blockIdx.y];
    
    // Shared memory for efficient accumulation
    __shared__ float shared_A[27];
    if (threadIdx.x < 27) shared_A[threadIdx.x] = 0.0f;
    __syncthreads();

    int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x;
    while (in_ind < blockDim.x * (blockIdx.x + 1) * n_accum && in_ind < n_disparity) {

        // fetch disparity, Zbuffer and normal from global memory
        float disp = d_disparity_compact[in_ind + start_ind];
        float4 tmp = d_Zbuffer_normals_compact[in_ind + start_ind];
        float Zbuffer = tmp.x;
        float nx = tmp.y;
        float ny = tmp.z;
        float nz = tmp.w;

        // compute coordinates
        int pixel_ind = d_ind_disparity_Zbuffer[in_ind + start_ind];

        float y = floorf(__fdividef((float)pixel_ind, n_cols));
        float x = (float)pixel_ind - y * n_cols;

        x = __fdividef((x - ox), fx);
        y = __fdividef((y - oy), fy);

        // reconstruct 3D point from disparity
        float Zd = -(fx * b) / disp;
        float Xd = x * Zd;
        float Yd = y * Zd;

        // reconstruct 3D point from model
        float Zm = Zbuffer;
        float Xm = x * Zm;
        float Ym = y * Zm;

        // weight the constraint according to (fx*b)/(Zm*Zm)
        float w2 = fx * b / (Zm * Zm);
        w2 *= w2;

        /************************/
        /* evaluate constraints */
        /************************/

        atomicAdd(&shared_A[0], w2 * (nx * nx));
        atomicAdd(&shared_A[1], w2 * (nx * ny));
        atomicAdd(&shared_A[2], w2 * (nx * nz));
        atomicAdd(&shared_A[3], w2 * (Ym * nx * nz - Zm * nx * ny));
        atomicAdd(&shared_A[4], w2 * (Zm * (nx * nx) - Xm * nx * nz));
        atomicAdd(&shared_A[5], w2 * (-Ym * (nx * nx) + Xm * nx * ny));

        atomicAdd(&shared_A[6], w2 * (ny * ny));
        atomicAdd(&shared_A[7], w2 * (ny * nz));
        atomicAdd(&shared_A[8], w2 * (-Zm * (ny * ny) + Ym * ny * nz));
        atomicAdd(&shared_A[9], w2 * (-Xm * ny * nz + Zm * nx * ny));
        atomicAdd(&shared_A[10], w2 * (Xm * (ny * ny) - Ym * nx * ny));

        atomicAdd(&shared_A[11], w2 * (nz * nz));
        atomicAdd(&shared_A[12], w2 * (Ym * (nz * nz) - Zm * ny * nz));
        atomicAdd(&shared_A[13], w2 * (-Xm * (nz * nz) + Zm * nx * nz));
        atomicAdd(&shared_A[14], w2 * (Xm * ny * nz - Ym * nx * nz));

        atomicAdd(&shared_A[15], w2 * ((Ym * Ym) * (nz * nz) + (Zm * Zm) * (ny * ny) - Ym * Zm * ny * nz * 2.0f));
        atomicAdd(&shared_A[16], w2 * (-Xm * Ym * (nz * nz) - (Zm * Zm) * nx * ny + Xm * Zm * ny * nz + Ym * Zm * nx * nz));
        atomicAdd(&shared_A[17], w2 * (-Xm * Zm * (ny * ny) - (Ym * Ym) * nx * nz + Xm * Ym * ny * nz + Ym * Zm * nx * ny));

        atomicAdd(&shared_A[18], w2 * ((Xm * Xm) * (nz * nz) + (Zm * Zm) * (nx * nx) - Xm * Zm * nx * nz * 2.0f));
        atomicAdd(&shared_A[19], w2 * (-Ym * Zm * (nx * nx) - (Xm * Xm) * ny * nz + Xm * Ym * nx * nz + Xm * Zm * nx * ny));

        atomicAdd(&shared_A[20], w2 * ((Xm * Xm) * (ny * ny) + (Ym * Ym) * (nx * nx) - Xm * Ym * nx * ny * 2.0f));

        // B-vector
        atomicAdd(&shared_A[21], w2 * (Xd * (nx * nx) - Xm * (nx * nx) + Yd * nx * ny - Ym * nx * ny + Zd * nx * nz - Zm * nx * nz));
        atomicAdd(&shared_A[22], w2 * (Yd * (ny * ny) - Ym * (ny * ny) + Xd * nx * ny - Xm * nx * ny + Zd * ny * nz - Zm * ny * nz));
        atomicAdd(&shared_A[23], w2 * (Zd * (nz * nz) - Zm * (nz * nz) + Xd * nx * nz - Xm * nx * nz + Yd * ny * nz - Ym * ny * nz));
        atomicAdd(&shared_A[24], w2 * (-Yd * Zm * (ny * ny) + Ym * Zd * (nz * nz) + Ym * Zm * (ny * ny) - Ym * Zm * (nz * nz) - (Ym * Ym) * ny * nz + (Zm * Zm) * ny * nz + Xd * Ym * nx * nz - Xm * Ym * nx * nz - Xd * Zm * nx * ny + Yd * Ym * ny * nz + Xm * Zm * nx * ny - Zd * Zm * ny * nz));
        atomicAdd(&shared_A[25], w2 * (Xd * Zm * (nx * nx) - Xm * Zd * (nz * nz) - Xm * Zm * (nx * nx) + Xm * Zm * (nz * nz) + (Xm * Xm) * nx * nz - (Zm * Zm) * nx * nz - Xd * Xm * nx * nz - Xm * Yd * ny * nz + Xm * Ym * ny * nz + Yd * Zm * nx * ny - Ym * Zm * nx * ny + Zd * Zm * nx * nz));
        atomicAdd(&shared_A[26], w2 * (-Xd * Ym * (nx * nx) + Xm * Yd * (ny * ny) + Xm * Ym * (nx * nx) - Xm * Ym * (ny * ny) - (Xm * Xm) * nx * ny + (Ym * Ym) * nx * ny + Xd * Xm * nx * ny - Yd * Ym * nx * ny + Xm * Zd * ny * nz - Xm * Zm * ny * nz - Ym * Zd * nx * nz + Ym * Zm * nx * nz));

        in_ind += blockDim.x;
    }
    __syncthreads();

    // Only a single thread writes the output from shared memory to global
    if (threadIdx.x < 27) {
        int out_ind = 27 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
        w_disp *= w_disp; // weight relative to flow
        d_CD[out_ind] = w_disp * shared_A[threadIdx.x];
    }
}