#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertFloatToRGBAbinary_kernel(uchar4 *out_image, const float *in_image, int width, int height, float lowerLim, float upperLim) {
    // Calculate the index using modern syntax and avoiding deprecated __mul24
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int index = y * width + x;
        float val = in_image[index];
        uchar4 temp;

        // Use single condition to initialize temp directly
        if (!isfinite(val) || val < lowerLim || val > upperLim) {
            temp = make_uchar4(255, 255, 255, 255); // White color
        } else {
            temp = make_uchar4(0, 0, 0, 0); // Black color
        }
        out_image[index] = temp;
    }
}