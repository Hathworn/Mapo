#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void createAnaglyph_kernel(uchar4 *out_image, const float *left_image, const float *right_image, int width, int height, int pre_shift) {
    // Using built-in multiplication function to avoid __mul24 deprecation
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int x_right = x - pre_shift;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) {
        uchar4 temp;
        
        temp.x = left_image[y * width + x];
        
        if (x_right >= 0 && x_right < width) {
            temp.y = right_image[y * width + x_right];
            temp.z = temp.y;
        } else {
            temp.y = 0;
            temp.z = 0;
        }
        
        temp.w = 255;
        
        // Store result in output image
        out_image[y * width + x] = temp;
    }
}