#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define IDX(x, y, width) ((y) * (width) + (x))

__global__ void convertFloatToRGBA_kernel(uchar4 *out_image, const float *in_image, int width, int height, float lowerLim, float upperLim) {
    // Calculate 1D index within the current block based on linearized thread index
    int idx = IDX(__mul24(blockIdx.y, blockDim.y) + threadIdx.y, 
                  __mul24(blockIdx.x, blockDim.x) + threadIdx.x, width);

    // Check if idx is within image bounds
    if (idx >= width * height) return;

    uchar4 temp;
    float val = in_image[idx];

    if (!isfinite(val)) {  // If value is not finite, set to white
        temp = make_uchar4(255, 255, 255, 255);
    } else {
        // Normalize value from [lowerLim, upperLim] to [0, 1]
        val = (val - lowerLim) / (upperLim - lowerLim);

        float r = 1.0f, g = 1.0f, b = 1.0f;
        // Map value to RGB using conditional ranges
        if (val < 0.25f) {
            r = 0;
            g = 4.0f * val;
        } else if (val < 0.5f) {
            r = 0;
            b = 1.0 + 4.0f * (0.25f - val);
        } else if (val < 0.75f) {
            r = 4.0f * (val - 0.5f);
            b = 0;
        } else {
            g = 1.0f + 4.0f * (0.75f - val);
            b = 0;
        }
        temp = make_uchar4(255.0 * r, 255.0 * g, 255.0 * b, 255);
    }

    out_image[idx] = temp;  // Assign computed color to output
}