#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertKinectFloatToRGBA_kernel(uchar4 *out_image, const float *in_image, int width, int height, int pitch, float lowerLim, float upperLim) {
    // Calculate pixel position
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check pixel boundaries
    if (x < width && y < height) {
        uchar4 temp;
        float val = *((float *)((char *)in_image + y * pitch) + x); // Access image value

        // Handle nan values
        if (val == 0.0f || !isfinite(val)) {
            temp = make_uchar4(255, 255, 255, 255);
        } else {
            // Rescale value from [lowerLim, upperLim] to [0, 1]
            val = (val - lowerLim) / (upperLim - lowerLim);

            float r = 1.0f, g = 1.0f, b = 1.0f;
            if (val < 0.25f) {
                r = 0;
                g = 4.0f * val;
            } else if (val < 0.5f) {
                r = 0;
                b = 1.0f + 4.0f * (0.25f - val);
            } else if (val < 0.75f) {
                r = 4.0f * (val - 0.5f);
                b = 0;
            } else {
                g = 1.0f + 4.0f * (0.75f - val);
                b = 0;
            }
            temp = make_uchar4(static_cast<unsigned char>(255.0f * r),
                               static_cast<unsigned char>(255.0f * g),
                               static_cast<unsigned char>(255.0f * b), 255);
        }
        out_image[y * width + x] = temp; // Assign to output image
    }
}