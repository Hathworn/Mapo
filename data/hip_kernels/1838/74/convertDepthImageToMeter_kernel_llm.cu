#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertDepthImageToMeter_kernel(float *d_depth_image_meter, const unsigned int *d_depth_image_millimeter, int n_rows, int n_cols) {
    // Calculate global thread index
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check bounds
    if (x < n_cols && y < n_rows) {
        int ind = y * n_cols + x;
        unsigned int depth = d_depth_image_millimeter[ind];
        
        // Check for invalid depth and conversion
        d_depth_image_meter[ind] = (depth == 0xFFFFFFFF) ? nanf("") : static_cast<float>(depth) * 0.001f;
    }
}