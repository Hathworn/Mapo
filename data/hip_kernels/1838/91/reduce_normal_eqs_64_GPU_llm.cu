#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_normal_eqs_64_GPU(float *d_C_reduced, float *d_C, int gridDim_x_normal_equations) {
    int tid = threadIdx.x;
    int bx = blockIdx.x;
    int ind = blockIdx.y * gridDim.x * gridDim_x_normal_equations * 64 +
              bx * gridDim_x_normal_equations * 64 + tid;

    __shared__ float DATA[64];

    // load and sum the first 20 elements
    float tmp = 0.0f;
    for (int i = 0; i < gridDim_x_normal_equations; i++)
        tmp += d_C[ind + i * 64];
    DATA[tid] = tmp;

    __syncthreads(); // ensure reading stage has finished

    // optimize using warp-shuffle for reduction
    for (int offset = 32; offset > 0; offset /= 2) {
        tmp += __shfl_down_sync(0xFFFFFFFF, tmp, offset);
    }

    // write results
    if (tid == 0)
        d_C_reduced[blockIdx.y * gridDim.x + bx] = tmp;
}