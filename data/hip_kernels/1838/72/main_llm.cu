#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "convertPointCloudToDepthImage_kernel_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    unsigned int *depth_image = NULL;
hipMalloc(&depth_image, XSIZE*YSIZE);
const float4 *point_cloud = NULL;
hipMalloc(&point_cloud, XSIZE*YSIZE);
int n_cols = 1;
int n_rows = 1;
int n_points = 1;
float nodal_point_x = 1;
float nodal_point_y = 1;
float focal_length_x = 1;
float focal_length_y = 1;
const float *T = NULL;
hipMalloc(&T, XSIZE*YSIZE);
const float *R = NULL;
hipMalloc(&R, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
convertPointCloudToDepthImage_kernel<<<gridBlock, threadBlock>>>(depth_image,point_cloud,n_cols,n_rows,n_points,nodal_point_x,nodal_point_y,focal_length_x,focal_length_y,T,R);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
convertPointCloudToDepthImage_kernel<<<gridBlock, threadBlock>>>(depth_image,point_cloud,n_cols,n_rows,n_points,nodal_point_x,nodal_point_y,focal_length_x,focal_length_y,T,R);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
convertPointCloudToDepthImage_kernel<<<gridBlock, threadBlock>>>(depth_image,point_cloud,n_cols,n_rows,n_points,nodal_point_x,nodal_point_y,focal_length_x,focal_length_y,T,R);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}