#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convertPitchedFloatToGrayRGBA_kernel(uchar4 *out_image, const float *in_image, int width, int height, int pitch, float lowerLim, float upperLim) {
    // Calculate global thread index using built-in functions
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        // Compute input index and access float value directly
        float val = *((float *)((char *)in_image + y * pitch) + x);

        // Rescale value from [lowerLim, upperLim] to [0, 255]
        val = 255.0f * (val - lowerLim) / (upperLim - lowerLim);
        val = max(0.0f, min(255.0f, val)); // Clamp to [0, 255]

        uchar4 temp;
        temp.x = (unsigned char)val;
        temp.y = (unsigned char)val;
        temp.z = (unsigned char)val;
        temp.w = 255;

        // Output index calculation using faster arithmetic
        out_image[y * width + x] = temp;
    }
}