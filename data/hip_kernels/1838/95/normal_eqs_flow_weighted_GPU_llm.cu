#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ static float flow_absolute_residual(float x, float y, float ux, float uy, float d, float fx, float fy, float T0, float T1, float T2, float R0, float R1, float R2) {
    float rx = -ux + fx * R1 - y * R2 + ((x * x) * R1) / fx + d * fx * T0 -
               d * x * T2 - (x * y * R0) / fx;
    float ry = -uy - fy * R0 + x * R2 - d * y * T2 - ((y * y) * R0) / fy +
               d * fy * T1 + (x * y * R1) / fy;

    return sqrtf(rx * rx + ry * ry);
}

__global__ void normal_eqs_flow_weighted_GPU(float *d_CO, const float2 *d_flow_compact, const float *d_Zbuffer_flow_compact, const int *d_ind_flow_Zbuffer, float fx, float fy, float ox, float oy, int n_rows, int n_cols, const int *d_n_values_flow, const int *d_start_ind_flow, const float *d_abs_res_scales, float w_flow, float w_ar_flow, const float *d_dTR) {

    int n_val_accum = gridDim.x * blockDim.x; // _MAX_N_VAL_ACCUM may not be multiple of blocksize

    int n_flow = d_n_values_flow[blockIdx.y];
    int n_accum = (int)ceilf((float)n_flow / (float)n_val_accum);
    int start_ind = d_start_ind_flow[blockIdx.y];

    // initialize accumulators
    float A[23] = {0.0f};

    for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x; in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {

        if (in_ind < n_flow) { // is this a valid sample?

            // fetch flow and Zbuffer from global memory
            float2 u = d_flow_compact[in_ind + start_ind];
            float disp = __fdividef(1.0f, d_Zbuffer_flow_compact[in_ind + start_ind]);

            // compute coordinates
            int pixel_ind = d_ind_flow_Zbuffer[in_ind + start_ind];
            bool is_ar_flow = (pixel_ind >= (n_rows * n_cols));
            pixel_ind -= (int)is_ar_flow * n_rows * n_cols;

            float y = floorf(__fdividef((float)pixel_ind, n_cols));
            float x = (float)pixel_ind - y * n_cols;

            x = x - ox;
            y = y - oy;

            // determine M-estimation weight
            float w_rel = is_ar_flow ? w_ar_flow : w_flow;
            int s6 = blockIdx.y * 6;
            float w = w_rel * flow_absolute_residual(x, y, u.x, u.y, disp, fx, fy, d_dTR[s6], d_dTR[s6 + 1], d_dTR[s6 + 2], d_dTR[s6 + 3], d_dTR[s6 + 4], d_dTR[s6 + 5]);
            w /= d_abs_res_scales[blockIdx.y];
            w = (w > 1) ? 0 : (1.0f - 2.0f * w * w + w * w * w * w);

            /************************/
            /* evaluate constraints */
            /************************/

            A[0] += w * (disp * disp * fx * fx);
            A[1] += w * (-disp * disp * x * fx);
            A[2] += w * (-disp * x * y);
            A[3] += w * (disp * fx * fx + disp * x * x);
            A[4] += w * (-disp * y * fx);
            A[5] += w * (-disp * disp * y * fy);
            A[6] += w * (-disp * fy * fy - disp * y * y);
            A[7] += w * (disp * x * fy);
            A[8] += w * (disp * disp * x * x + disp * disp * y * y);
            A[9] += w * (disp * x * x * y / fx + disp * y * fy + disp * y * y * y / fy);
            A[10] += w * (-disp * x * fx - disp * x * x * x / fx - disp * x * y * y / fy);
            A[11] += w * (x * x * y * y / (fx * fx) + fy * fy + 2.0f * y * y + y * y * y * y / (fy * fy));
            A[12] += w * (-2.0f * x * y - x * x * x * y / (fx * fx) - x * y * y * y / (fy * fy));
            A[13] += w * (x * y * y / fx - x * fy - x * y * y / fy);
            A[14] += w * (fx * fx + 2.0f * x * x + x * x * x * x / (fx * fx) + x * x * y * y / (fy * fy));
            A[15] += w * (-y * fx - x * x * y / fx + x * x * y / fy);
            A[16] += w * (x * x + y * y);

            // B-vector
            A[17] += w * (disp * u.x * fx);
            A[18] += w * (disp * u.y * fy);
            A[19] += w * (-disp * x * u.x - disp * y * u.y);
            A[20] += w * (-x * y * u.x / fx - u.y * fy - u.y * y * y / fy);
            A[21] += w * (u.x * fx + x * x * u.x / fx + x * y * u.y / fy);
            A[22] += w * (-y * u.x + x * u.y);
        }
    }

    /**************************/
    /* write out accumulators */
    /**************************/

    int out_ind = 23 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;

    #pragma unroll
    for (int i = 0; i < 23; ++i) {
        d_CO[out_ind + i * n_val_accum] = A[i];
    }
}