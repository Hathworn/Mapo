#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void colorInvalids_kernel(uchar4 *out_image, const float *in_image, int width, int height) {
    // Calculate global index using faster dim3 multiplication
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int ind = y * width + x; // Simplified multiplication for better readability
        uchar4 temp = out_image[ind];
        float value = in_image[ind];

        if (!isfinite(value)) {
            // Efficient operation by pre-computing factor
            float factor = 0.5f;
            temp.x *= factor;
            temp.y *= factor;
        }

        out_image[ind] = temp; // Move assignment outside conditional
    }
}