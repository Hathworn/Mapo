#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void median_reduce_shuffle_gpu(const float *d_in, float *d_out, float *d_random_numbers, int n_in) {

    // Compute global index
    int g_ind = blockIdx.x * MED_BLOCK_SIZE + threadIdx.x;

    // Allocate shared memory
    extern __shared__ float DATA[];

    // Load stage with random sampling
    int sample_ind = floorf(d_random_numbers[g_ind] * (float)n_in);
    DATA[threadIdx.x] = d_in[sample_ind];

    __syncthreads();

    /*******************/
    /* Reduction stage */
    /*******************/

    for (int s = 1; s < MED_BLOCK_SIZE; s *= 3) {

        int index = 3 * s * threadIdx.x;

        if (index + 2 * s < MED_BLOCK_SIZE) { // Ensure valid range for fetching values

            // Fetch three values
            float value1 = DATA[index];
            float value2 = DATA[index + s];
            float value3 = DATA[index + 2 * s];

            // Extract the middle value (median)
            float smallest = fminf(value1, value2);
            value2 = fmaxf(value1, value2);
            value1 = smallest;

            value3 = fmaxf(value1, value3);
            value2 = fminf(value2, value3);

            DATA[index] = value2;
        }

        __syncthreads();
    }

    // Write stage
    if (threadIdx.x == 0) {
        d_out[blockIdx.x] = DATA[0];
    }
}