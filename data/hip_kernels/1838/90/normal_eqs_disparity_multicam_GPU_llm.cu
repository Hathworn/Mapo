#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normal_eqs_disparity_multicam_GPU(float *d_CD, float *d_disparity_compact, float4 *d_Zbuffer_normals_compact, int *d_ind_disparity_Zbuffer, const float *d_focal_length, const float *d_nodal_point_x, const float *d_nodal_point_y, const float *d_baseline, const int *d_n_cols, const int *d_n_values_disparity, const int *d_start_ind_disparity, const int *d_pixel_ind_offset) {
    int n_val_accum = gridDim.x * blockDim.x;

    int n_disparity = d_n_values_disparity[blockIdx.y];
    int n_accum = (int)ceilf((float)n_disparity / (float)n_val_accum);
    int start_ind = d_start_ind_disparity[blockIdx.y];

    float f = d_focal_length[blockIdx.y];
    float ox = d_nodal_point_x[blockIdx.y];
    float oy = d_nodal_point_y[blockIdx.y];
    float b = d_baseline[blockIdx.y];
    int n_cols = d_n_cols[blockIdx.y];
    int pixel_ind_offset = d_pixel_ind_offset[blockIdx.y];

    // Initialize accumulators
    float A[27] = {0.0f};

    for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x;
         in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {

        if (in_ind < n_disparity) {
            // Fetch disparity, Zbuffer and normal from global memory
            float disp = d_disparity_compact[in_ind + start_ind];
            float4 tmp = d_Zbuffer_normals_compact[in_ind + start_ind];
            float Zbuffer = tmp.x;
            float nx = tmp.y;
            float ny = tmp.z;
            float nz = tmp.w;

            // Compute coordinates
            int pixel_ind = d_ind_disparity_Zbuffer[in_ind + start_ind] - pixel_ind_offset;

            float y = floorf(__fdividef((float)pixel_ind, n_cols));
            float x = (float)pixel_ind - y * n_cols;

            x = __fdividef((x - ox), f);
            y = -__fdividef((y - oy), f);

            // Reconstruct 3D point from disparity
            float Zd = -(f * b) / disp;
            float Xd = x * Zd;
            float Yd = y * Zd;

            // Reconstruct 3D point from model
            float Zm = Zbuffer;
            float Xm = x * Zm;
            float Ym = y * Zm;

            // Evaluate constraints and update unique values in A-matrix
            A[0] += nx * nx;
            A[1] += nx * ny;
            A[2] += nx * nz;
            A[3] += Ym * nx * nz - Zm * nx * ny;
            A[4] += Zm * (nx * nx) - Xm * nx * nz;
            A[5] += -Ym * (nx * nx) + Xm * nx * ny;

            A[6] += ny * ny;
            A[7] += ny * nz;
            A[8] += -Zm * (ny * ny) + Ym * ny * nz;
            A[9] += -Xm * ny * nz + Zm * nx * ny;
            A[10] += Xm * (ny * ny) - Ym * nx * ny;

            A[11] += nz * nz;
            A[12] += Ym * (nz * nz) - Zm * ny * nz;
            A[13] += -Xm * (nz * nz) + Zm * nx * nz;
            A[14] += Xm * ny * nz - Ym * nx * nz;

            A[15] += (Ym * Ym) * (nz * nz) + (Zm * Zm) * (ny * ny) - Ym * Zm * ny * nz * 2.0f;
            A[16] += -Xm * Ym * (nz * nz) - (Zm * Zm) * nx * ny + Xm * Zm * ny * nz + Ym * Zm * nx * nz;
            A[17] += -Xm * Zm * (ny * ny) - (Ym * Ym) * nx * nz + Xm * Ym * ny * nz + Ym * Zm * nx * ny;

            A[18] += (Xm * Xm) * (nz * nz) + (Zm * Zm) * (nx * nx) - Xm * Zm * nx * nz * 2.0f;
            A[19] += -Ym * Zm * (nx * nx) - (Xm * Xm) * ny * nz + Xm * Ym * nx * nz + Xm * Zm * nx * ny;

            A[20] += (Xm * Xm) * (ny * ny) + (Ym * Ym) * (nx * nx) - Xm * Ym * nx * ny * 2.0f;

            // Update B-vector
            A[21] += Xd * (nx * nx) - Xm * (nx * nx) + Yd * nx * ny - Ym * nx * ny + Zd * nx * nz - Zm * nx * nz;
            A[22] += Yd * (ny * ny) - Ym * (ny * ny) + Xd * nx * ny - Xm * nx * ny + Zd * ny * nz - Zm * ny * nz;
            A[23] += Zd * (nz * nz) - Zm * (nz * nz) + Xd * nx * nz - Xm * nx * nz + Yd * ny * nz - Ym * ny * nz;
            A[24] += -Yd * Zm * (ny * ny) + Ym * Zd * (nz * nz) + Ym * Zm * (ny * ny) - Ym * Zm * (nz * nz) - (Ym * Ym) * ny * nz + (Zm * Zm) * ny * nz +
                       Xd * Ym * nx * nz - Xm * Ym * nx * nz - Xd * Zm * nx * ny + Yd * Ym * ny * nz + Xm * Zm * nx * ny - Zd * Zm * ny * nz;
            A[25] += Xd * Zm * (nx * nx) - Xm * Zd * (nz * nz) - Xm * Zm * (nx * nx) + Xm * Zm * (nz * nz) + (Xm * Xm) * nx * nz - (Zm * Zm) * nx * nz -
                       Xd * Xm * nx * nz - Xm * Yd * ny * nz + Xm * Ym * ny * nz + Yd * Zm * nx * ny - Ym * Zm * nx * ny + Zd * Zm * nx * nz;
            A[26] += -Xd * Ym * (nx * nx) + Xm * Yd * (ny * ny) + Xm * Ym * (nx * nx) - Xm * Ym * (ny * ny) - (Xm * Xm) * nx * ny + (Ym * Ym) * nx * ny +
                       Xd * Xm * nx * ny - Yd * Ym * nx * ny + Xm * Zd * ny * nz - Xm * Zm * ny * nz - Ym * Zd * nx * nz + Ym * Zm * nx * nz;
        }
    }

    // Write out accumulators
    int out_ind = 27 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = 0; i < 27; ++i) {
        d_CD[out_ind + i * n_val_accum] = A[i];
    }
}