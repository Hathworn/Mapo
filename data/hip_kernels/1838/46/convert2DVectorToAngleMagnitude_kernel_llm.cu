#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convert2DVectorToAngleMagnitude_kernel(uchar4 *d_angle_image, uchar4 *d_magnitude_image, float *d_vector_X, float *d_vector_Y, int width, int height, float lower_ang, float upper_ang, float lower_mag, float upper_mag) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int idx = y * width + x;
        float vector_X = d_vector_X[idx];
        float vector_Y = d_vector_Y[idx];

        // compute angle and magnitude
        float angle = atan2f(vector_Y, vector_X);
        float magnitude = hypotf(vector_X, vector_Y);

        // prepare output colors
        uchar4 temp_angle;
        uchar4 temp_magnitude;

        if (!isfinite(magnitude)) {
            // Unmatched pixels in white
            temp_angle = make_uchar4(255, 255, 255, 255);
            temp_magnitude = make_uchar4(255, 255, 255, 255);
        } else {
            // rescale angle and magnitude
            angle = (angle - lower_ang) / (upper_ang - lower_ang);
            magnitude = (magnitude - lower_mag) / (upper_mag - lower_mag);

            auto calculate_color = [](float value) -> uchar4 {
                float r = 1.0f, g = 1.0f, b = 1.0f;
                if (value < 0.25f) {
                    r = 0; g = 4.0f * value;
                } else if (value < 0.5f) {
                    r = 0; b = 1.0f + 4.0f * (0.25f - value);
                } else if (value < 0.75f) {
                    r = 4.0f * (value - 0.5f); b = 0;
                } else {
                    g = 1.0f + 4.0f * (0.75f - value); b = 0;
                }
                return make_uchar4(255.0f * r, 255.0f * g, 255.0f * b, 255);
            };

            temp_angle = calculate_color(angle);
            temp_magnitude = calculate_color(magnitude);
        }

        // store the results
        d_angle_image[idx] = temp_angle;
        d_magnitude_image[idx] = temp_magnitude;
    }
}