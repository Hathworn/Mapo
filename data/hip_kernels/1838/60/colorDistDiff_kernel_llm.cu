#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void colorDistDiff_kernel(uchar4 *out_image, const float *disparity, int disparity_pitch, const float *disparity_prior, int width, int height, float f, float b, float ox, float oy, float dist_thres) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int ind = y * width + x;
        uchar4 temp = out_image[ind];
        float disp = *((float *)((char *)disparity + y * disparity_pitch) + x);
        float disp_model = disparity_prior[ind];

        // Precompute reciprocal for optimization
        float recip_f = __frcp_rn(f);
        
        // 3D reconstruct and measure Euclidian distance
        float xt = (x - ox) * recip_f;
        float yt = -(y - oy) * recip_f; // coord. transform

        float Zm = -(f * b) * __frcp_rn(disp_model);
        float Xm = xt * Zm;
        float Ym = yt * Zm;

        float Zd = -(f * b) * __frcp_rn(disp);
        float Xd = xt * Zd;
        float Yd = yt * Zd;

        float d_md = sqrtf((Xm - Xd) * (Xm - Xd) + (Ym - Yd) * (Ym - Yd) + (Zm - Zd) * (Zm - Zd));

        bool color = (d_md > dist_thres) || (isfinite(disp) && !isfinite(disp_model));

        if (color) { // color
            temp.x *= 0.5f;
            temp.y *= 0.5f;
        }

        out_image[ind] = temp;
    }
}