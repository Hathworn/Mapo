#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "flow_absolute_residual_scalable_GPU.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *d_abs_res = NULL;
hipMalloc(&d_abs_res, XSIZE*YSIZE);
const float2 *d_flow_compact = NULL;
hipMalloc(&d_flow_compact, XSIZE*YSIZE);
const float *d_Zbuffer_flow_compact = NULL;
hipMalloc(&d_Zbuffer_flow_compact, XSIZE*YSIZE);
const int *d_ind_flow_Zbuffer = NULL;
hipMalloc(&d_ind_flow_Zbuffer, XSIZE*YSIZE);
const unsigned int *d_valid_flow_Zbuffer = NULL;
hipMalloc(&d_valid_flow_Zbuffer, XSIZE*YSIZE);
float fx = 1;
float fy = 1;
float ox = 1;
float oy = 1;
int n_rows = 1;
int n_cols = 1;
int n_valid_flow_Zbuffer = 1;
const int *d_offset_ind = NULL;
hipMalloc(&d_offset_ind, XSIZE*YSIZE);
const int *d_segment_translation_table = NULL;
hipMalloc(&d_segment_translation_table, XSIZE*YSIZE);
float w_flow = 1;
float w_ar_flow = 1;
const float *d_dTR = NULL;
hipMalloc(&d_dTR, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
flow_absolute_residual_scalable_GPU<<<gridBlock, threadBlock>>>(d_abs_res,d_flow_compact,d_Zbuffer_flow_compact,d_ind_flow_Zbuffer,d_valid_flow_Zbuffer,fx,fy,ox,oy,n_rows,n_cols,n_valid_flow_Zbuffer,d_offset_ind,d_segment_translation_table,w_flow,w_ar_flow,d_dTR);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
flow_absolute_residual_scalable_GPU<<<gridBlock, threadBlock>>>(d_abs_res,d_flow_compact,d_Zbuffer_flow_compact,d_ind_flow_Zbuffer,d_valid_flow_Zbuffer,fx,fy,ox,oy,n_rows,n_cols,n_valid_flow_Zbuffer,d_offset_ind,d_segment_translation_table,w_flow,w_ar_flow,d_dTR);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
flow_absolute_residual_scalable_GPU<<<gridBlock, threadBlock>>>(d_abs_res,d_flow_compact,d_Zbuffer_flow_compact,d_ind_flow_Zbuffer,d_valid_flow_Zbuffer,fx,fy,ox,oy,n_rows,n_cols,n_valid_flow_Zbuffer,d_offset_ind,d_segment_translation_table,w_flow,w_ar_flow,d_dTR);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}