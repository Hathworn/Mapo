#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void invalidateFlow_kernel(float *modFlowX, float *modFlowY, const float *constFlowX, const float *constFlowY, int width, int height, float cons_thres) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int ind = y * width + x;  // Simplified index calculation
        float mFX = modFlowX[ind];
        float mFY = modFlowY[ind];
        float cFX = constFlowX[ind];
        float cFY = constFlowY[ind];

        // Direct computation without intermediate err variable
        float err = hypotf((mFX - cFX), (mFY - cFY));  // Use hypotf for efficient calculation

        if (err > cons_thres) {
            mFX = nanf("");
            mFY = nanf("");
        }

        modFlowX[ind] = mFX;
        modFlowY[ind] = mFY;
    }
}