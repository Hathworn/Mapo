#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample_ind_and_labels_GPU(int *d_ind_sub, const int *d_ind, unsigned int *d_label_sub, const unsigned int *d_label, int n_out, float inv_sub_factor) {

    unsigned int ind_out = blockIdx.x * blockDim.x + threadIdx.x;

    if (ind_out < n_out) {
        // Use __fmul_rn for fast and precise multiplication
        int ind_in = __fmul_rn(ind_out, inv_sub_factor);
        d_ind_sub[ind_out] = __ldg(&d_ind[ind_in]); // Leverage __ldg for read-only cache
        d_label_sub[ind_out] = __ldg(&d_label[ind_in]); // Leverage __ldg for read-only cache
    }
}