#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertKinectDisparityInPlace_kernel(float *d_disparity, int pitch, int width, int height, float depth_scale) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return; // Exit early if out of bounds

    float *d_in = (float *)((char *)d_disparity + y * pitch) + x;
    float value = *d_in;
    *d_in = (value == 0.0f) ? nanf("") : (-depth_scale / value); // Precompute value and remove redundant check
}