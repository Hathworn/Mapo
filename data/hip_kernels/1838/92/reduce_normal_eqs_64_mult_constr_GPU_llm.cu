#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_normal_eqs_64_mult_constr_GPU(float *d_C_reduced, const float *d_C, int gridDim_x_normal_equations, int n_constraints) {
    // Check if there are constraints left to be processed
    int constraint_ind = blockIdx.x * 4 + threadIdx.y;

    if (constraint_ind < n_constraints) {
        int tid = 64 * threadIdx.y + threadIdx.x;

        // Put data in shared memory
        int ind = blockIdx.y * n_constraints * gridDim_x_normal_equations * 64 +
                  constraint_ind * gridDim_x_normal_equations * 64 + threadIdx.x;

        __shared__ float DATA[64 * 4];

        // Load and sum the first gridDim_x_normal_equations elements
        float tmp = 0.0f;
        for (int i = 0; i < gridDim_x_normal_equations; i++)
            tmp += d_C[ind + i * 64];
        DATA[tid] = tmp;

        __syncthreads(); // Ensure reading stage has finished

        // Optimize warp-level reduction with loop and shared memory
        for (int offset = 32; offset > 0; offset /= 2) {
            if ((tid - 64 * threadIdx.y) < offset)
                DATA[tid] += DATA[tid + offset];
            __syncthreads();
        }

        // Write results
        if (threadIdx.x == 0)
            d_C_reduced[blockIdx.y * n_constraints + constraint_ind] = DATA[tid];
    }
}