#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normal_eqs_flow_GPU(float *d_CO, const float2 *d_flow_compact, const float *d_Zbuffer_flow_compact, const int *d_ind_flow_Zbuffer, float fx, float fy, float ox, float oy, int n_rows, int n_cols, const int *d_n_values_flow, const int *d_start_ind_flow) {

    extern __shared__ float shared_CO[];

    // Calculate the number of values accumulated per block
    int n_val_accum = gridDim.x * blockDim.x;

    // Get number of flow values for this block and start index for flow data
    int n_flow = d_n_values_flow[blockIdx.y];
    int n_accum = (int)ceilf((float)n_flow / (float)n_val_accum);
    int start_ind = d_start_ind_flow[blockIdx.y];

    // Initialize local accumulators
    float A[23] = {0.0f};

    for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x;
         in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {

        if (in_ind < n_flow) {
            // Fetch flow and Zbuffer from global memory
            float2 u = d_flow_compact[in_ind + start_ind];
            float disp = __fdividef(1.0f, d_Zbuffer_flow_compact[in_ind + start_ind]);

            // Compute coordinates
            int pixel_ind = d_ind_flow_Zbuffer[in_ind + start_ind];
            bool is_ar_flow = (pixel_ind >= (n_rows * n_cols));
            pixel_ind -= (int)is_ar_flow * n_rows * n_cols;

            float y = floorf(__fdividef((float)pixel_ind, n_cols));
            float x = (float)pixel_ind - y * n_cols;

            x = x - ox;
            y = y - oy;

            // Evaluate constraints
            A[0] += (disp * disp * fx * fx);
            A[1] += (-disp * disp * x * fx);
            A[2] += (-disp * x * y);
            A[3] += (disp * fx * fx + disp * x * x);
            A[4] += (-disp * y * fx);
            A[5] += (-disp * disp * y * fy);
            A[6] += (-disp * fy * fy - disp * y * y);
            A[7] += (disp * x * fy);
            A[8] += (disp * disp * x * x + disp * disp * y * y);
            A[9] += (disp * x * x * y / fx + disp * y * fy + disp * y * y * y / fy);
            A[10] += (-disp * x * fx - disp * x * x * x / fx - disp * x * y * y / fy);
            A[11] += (x * x * y * y / (fx * fx) + fy * fy + 2.0f * y * y + y * y * y * y / (fy * fy));
            A[12] += (-2.0f * x * y - x * x * x * y / (fx * fx) - x * y * y * y / (fy * fy));
            A[13] += (x * y * y / fx - x * fy - x * y * y / fy);
            A[14] += (fx * fx + 2.0f * x * x + x * x * x * x / (fx * fx) + x * x * y * y / (fy * fy));
            A[15] += (-y * fx - x * x * y / fx + x * x * y / fy);
            A[16] += (x * x + y * y);
            A[17] += (disp * u.x * fx);
            A[18] += (disp * u.y * fy);
            A[19] += (-disp * x * u.x - disp * y * u.y);
            A[20] += (-x * y * u.x / fx - u.y * fy - u.y * y * y / fy);
            A[21] += (u.x * fx + x * x * u.x / fx + x * y * u.y / fy);
            A[22] += (-y * u.x + x * u.y);
        }
    }

    // Use shared memory to accumulate results
    for (int i = 0; i < 23; ++i) {
        atomicAdd(&shared_CO[i * n_val_accum + threadIdx.x], A[i]);
    }
    
    __syncthreads();

    // Write out accumulators to global memory
    int out_ind = 23 * n_val_accum * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = 0; i < 23; ++i) {
        d_CO[out_ind + i * n_val_accum] = shared_CO[i * n_val_accum + threadIdx.x];
    }
}