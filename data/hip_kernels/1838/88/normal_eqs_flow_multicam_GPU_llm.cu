#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normal_eqs_flow_multicam_GPU(float *d_CO, float2 *d_flow_compact, float *d_Zbuffer_flow_compact, int *d_ind_flow_Zbuffer, const float *d_focal_length, const float *d_nodal_point_x, const float *d_nodal_point_y, const int *d_n_rows, const int *d_n_cols, const int *d_n_values_flow, const int *d_start_ind_flow, const int *d_pixel_ind_offset) {
    extern __shared__ float shared_data[]; // Use shared memory for accumulators
    float *s_A = &shared_data[threadIdx.x * 23];

    // Initialize shared accumulators
    for (int i = 0; i < 23; ++i) {
        s_A[i] = 0.0f;
    }

    int n_val_accum = gridDim.x * blockDim.x;
    int n_flow = d_n_values_flow[blockIdx.y];
    int n_accum = (int)ceilf((float)n_flow / (float)n_val_accum);
    int start_ind = d_start_ind_flow[blockIdx.y];

    float f = d_focal_length[blockIdx.y];
    float ox = d_nodal_point_x[blockIdx.y];
    float oy = d_nodal_point_y[blockIdx.y];
    int n_rows = d_n_rows[blockIdx.y];
    int n_cols = d_n_cols[blockIdx.y];
    int pixel_ind_offset = d_pixel_ind_offset[blockIdx.y];

    for (int in_ind = blockDim.x * blockIdx.x * n_accum + threadIdx.x; in_ind < blockDim.x * (blockIdx.x + 1) * n_accum; in_ind += blockDim.x) {
        if (in_ind < n_flow) {
            float2 u = d_flow_compact[in_ind + start_ind];
            float disp = __fdividef(1.0f, d_Zbuffer_flow_compact[in_ind + start_ind]);

            int pixel_ind = d_ind_flow_Zbuffer[in_ind + start_ind] - pixel_ind_offset;
            bool is_ar_flow = (pixel_ind >= (n_rows * n_cols));
            pixel_ind -= (int)is_ar_flow * n_rows * n_cols;

            float y = floorf(__fdividef((float)pixel_ind, n_cols));
            float x = (float)pixel_ind - y * n_cols;

            x = x - ox;
            y = y - oy;

            y = -y;
            u.y = -u.y;

            s_A[0] += (disp * disp * f * f);
            s_A[1] += (-disp * disp * x * f);
            s_A[2] += (-disp * x * y);
            s_A[3] += (disp * f * f + disp * x * x);
            s_A[4] += (-disp * y * f);
            s_A[5] += (-disp * disp * y * f);
            s_A[6] += (-disp * f * f - disp * y * y);
            s_A[7] += (disp * x * f);
            s_A[8] += (disp * disp * x * x + disp * disp * y * y);
            s_A[9] += (disp * x * x * y / f + disp * y * f + disp * y * y * y / f);
            s_A[10] += (-disp * x * f - disp * x * x * x / f - disp * x * y * y / f);
            s_A[11] += (x * x * y * y / (f * f) + f * f + 2.0f * y * y + y * y * y * y / (f * f));
            s_A[12] += (-2.0f * x * y - x * x * x * y / (f * f) - x * y * y * y / (f * f));
            s_A[13] += (-x * f);
            s_A[14] += (f * f + 2.0f * x * x + x * x * x * x / (f * f) + x * x * y * y / (f * f));
            s_A[15] += (-y * f);
            s_A[16] += (x * x + y * y);
            s_A[17] += (disp * u.x * f);
            s_A[18] += (disp * u.y * f);
            s_A[19] += (-disp * x * u.x - disp * y * u.y);
            s_A[20] += (-x * y * u.x / f - u.y * f - u.y * y * y / f);
            s_A[21] += (u.x * f + x * x * u.x / f + x * y * u.y / f);
            s_A[22] += (-y * u.x + x * u.y);
        }
    }

    __syncthreads(); // Synchronize threads before writing back

    // Aggregate the data across the block and write back to global memory
    if (threadIdx.x == 0) {
        for (int i = 0; i < 23; ++i) {
            float sum = 0.0f;
            for (int j = 0; j < blockDim.x; ++j) {
                sum += shared_data[j * 23 + i];
            }
            d_CO[23 * n_val_accum * blockIdx.y + i * n_val_accum + blockDim.x * blockIdx.x + threadIdx.x] = sum;
        }
    }
}