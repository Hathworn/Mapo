#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "normal_eqs_flow_multicam_GPU.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *d_CO = NULL;
hipMalloc(&d_CO, XSIZE*YSIZE);
float2 *d_flow_compact = NULL;
hipMalloc(&d_flow_compact, XSIZE*YSIZE);
float *d_Zbuffer_flow_compact = NULL;
hipMalloc(&d_Zbuffer_flow_compact, XSIZE*YSIZE);
int *d_ind_flow_Zbuffer = NULL;
hipMalloc(&d_ind_flow_Zbuffer, XSIZE*YSIZE);
const float *d_focal_length = NULL;
hipMalloc(&d_focal_length, XSIZE*YSIZE);
const float *d_nodal_point_x = NULL;
hipMalloc(&d_nodal_point_x, XSIZE*YSIZE);
const float *d_nodal_point_y = NULL;
hipMalloc(&d_nodal_point_y, XSIZE*YSIZE);
const int *d_n_rows = NULL;
hipMalloc(&d_n_rows, XSIZE*YSIZE);
const int *d_n_cols = NULL;
hipMalloc(&d_n_cols, XSIZE*YSIZE);
const int *d_n_values_flow = NULL;
hipMalloc(&d_n_values_flow, XSIZE*YSIZE);
const int *d_start_ind_flow = NULL;
hipMalloc(&d_start_ind_flow, XSIZE*YSIZE);
const int *d_pixel_ind_offset = NULL;
hipMalloc(&d_pixel_ind_offset, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
normal_eqs_flow_multicam_GPU<<<gridBlock, threadBlock>>>(d_CO,d_flow_compact,d_Zbuffer_flow_compact,d_ind_flow_Zbuffer,d_focal_length,d_nodal_point_x,d_nodal_point_y,d_n_rows,d_n_cols,d_n_values_flow,d_start_ind_flow,d_pixel_ind_offset);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
normal_eqs_flow_multicam_GPU<<<gridBlock, threadBlock>>>(d_CO,d_flow_compact,d_Zbuffer_flow_compact,d_ind_flow_Zbuffer,d_focal_length,d_nodal_point_x,d_nodal_point_y,d_n_rows,d_n_cols,d_n_values_flow,d_start_ind_flow,d_pixel_ind_offset);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
normal_eqs_flow_multicam_GPU<<<gridBlock, threadBlock>>>(d_CO,d_flow_compact,d_Zbuffer_flow_compact,d_ind_flow_Zbuffer,d_focal_length,d_nodal_point_x,d_nodal_point_y,d_n_rows,d_n_cols,d_n_values_flow,d_start_ind_flow,d_pixel_ind_offset);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}