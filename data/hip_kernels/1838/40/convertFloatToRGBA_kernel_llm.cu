#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertFloatToRGBA_kernel(uchar4 *out_image, const float *in_image, int width, int height) {
    // Calculate global thread indices for the x and y dimensions
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure thread is within image bounds
    if (x < width && y < height) {
        int IND = y * width + x; 
        float val = in_image[IND]; 
        
        // Set RGBA values using the same float input
        out_image[IND] = make_uchar4(val, val, val, 255);
    }
}