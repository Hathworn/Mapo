#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ static float rgbaToGray(uchar4 rgba) {
    return (0.299f * (float)rgba.x + 0.587f * (float)rgba.y + 0.114f * (float)rgba.z);
}

__global__ void createAnaglyph_kernel(uchar4 *out_image, const uchar4 *left_image, const uchar4 *right_image, int width, int height, int pre_shift) {
    // Calculate the global x and y indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // If indices are within image bounds
    if (x < width && y < height) {
        // Use shared memory to reduce global memory accesses
        __shared__ uchar4 shared_left[1024];
        __shared__ uchar4 shared_right[1024];
        
        // Load left and right images into shared memory
        int idx = threadIdx.y * blockDim.x + threadIdx.x;
        shared_left[idx] = left_image[y * width + x];
        int x_right = x - pre_shift;
        if (x_right >= 0 && x_right < width) {
            shared_right[idx] = right_image[y * width + x_right];
        }

        __syncthreads();

        uchar4 temp;
        temp.x = rgbaToGray(shared_left[idx]);

        if (x_right >= 0 && x_right < width) {
            temp.y = rgbaToGray(shared_right[idx]);
            temp.z = temp.y;
        } else {
            temp.y = 0;
            temp.z = 0;
        }

        temp.w = 255;

        // Write back the result to global memory
        out_image[y * width + x] = temp;
    }
}