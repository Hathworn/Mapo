#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "createAnaglyph_kernel.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    uchar4 *out_image = NULL;
hipMalloc(&out_image, XSIZE*YSIZE);
const uchar4 *left_image = NULL;
hipMalloc(&left_image, XSIZE*YSIZE);
const uchar4 *right_image = NULL;
hipMalloc(&right_image, XSIZE*YSIZE);
int width = XSIZE;
int height = YSIZE;
int pre_shift = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
createAnaglyph_kernel<<<gridBlock, threadBlock>>>(out_image,left_image,right_image,width,height,pre_shift);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
createAnaglyph_kernel<<<gridBlock, threadBlock>>>(out_image,left_image,right_image,width,height,pre_shift);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
createAnaglyph_kernel<<<gridBlock, threadBlock>>>(out_image,left_image,right_image,width,height,pre_shift);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}