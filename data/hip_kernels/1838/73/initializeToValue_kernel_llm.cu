#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initializeToValue_kernel(unsigned int *data, unsigned int value, int width, int height) {
    // Calculate a unique global thread ID
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Use a 1D index for improved memory coalescing
    int index = idy * width + idx;

    // Only proceed if indices are within bounds
    if (idx < width && idy < height) {
        data[index] = value;
    }
}