#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blendFloatImageFloatLabelToRGBA_kernel(uchar4 *out_image, const float *in_image, const float *label, int width, int height, float lowerLim, float upperLim) {
    // Use efficient calculation for thread index
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int idx = y * width + x;  // Calculate index once
    
    if (x < width && y < height) {
        unsigned char img = static_cast<unsigned char>(0.5f * in_image[idx] + 128.0f);
        float val = label[idx];
        
        uchar4 temp;
        // Simplify blend conditions
        if (!isfinite(val) || (val < lowerLim) || (val > upperLim)) {
            temp = make_uchar4(img, img, img, 255);
        } else {
            temp = make_uchar4(static_cast<unsigned char>(0.6f * img), static_cast<unsigned char>(0.6f * img), img, 255);
        }
        
        out_image[idx] = temp;
    }
}