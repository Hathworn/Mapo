#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertFlowToRGBA_kernel(uchar4 *d_flowx_out, uchar4 *d_flowy_out, const float *d_flowx_in, const float *d_flowy_in, int width, int height, float lowerLim, float upperLim, float minMag) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Use direct arithmetic for indexing
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Use direct arithmetic for indexing
    uchar4 tempx, tempy;
    
    if (x < width && y < height) {
        float ux = d_flowx_in[y * width + x]; // Use direct arithmetic for indexing
        float uy = d_flowy_in[y * width + x]; // Use direct arithmetic for indexing

        float mag = ux * ux + uy * uy; // Avoid sqrtf and direct calculation if possible
        
        // Handle non-finite ux or low magnitude values
        if (!isfinite(ux) || (mag < minMag * minMag)) { // Avoid sqrtf by comparing squared magnitudes
            tempx = make_uchar4(255, 255, 255, 255); // Use make_uchar4 for compact assignment
            tempy = make_uchar4(255, 255, 255, 255); // Use make_uchar4 for compact assignment
        } else {
            // Rescale ux
            ux = (ux - lowerLim) / (upperLim - lowerLim);
            // Initialize RGB components
            float r, g, b;
            
            if (ux < 0.25f) {
                r = 0;
                g = 4.0f * ux;
                b = 1.0f;
            } else if (ux < 0.5f) {
                r = 0;
                g = 1.0f;
                b = 1.0f + 4.0f * (0.25f - ux);
            } else if (ux < 0.75f) {
                r = 4.0f * (ux - 0.5f);
                g = 1.0f;
                b = 0;
            } else {
                r = 1.0f;
                g = 1.0f + 4.0f * (0.75f - ux);
                b = 0;
            }
            
            tempx = make_uchar4(255.0f * r, 255.0f * g, 255.0f * b, 255); // Use make_uchar4 for compact assignment

            // Rescale uy
            uy = (uy - lowerLim) / (upperLim - lowerLim);
            
            if (uy < 0.25f) {
                r = 0;
                g = 4.0f * uy;
                b = 1.0f;
            } else if (uy < 0.5f) {
                r = 0;
                g = 1.0f;
                b = 1.0f + 4.0f * (0.25f - uy);
            } else if (uy < 0.75f) {
                r = 4.0f * (uy - 0.5f);
                g = 1.0f;
                b = 0;
            } else {
                r = 1.0f;
                g = 1.0f + 4.0f * (0.75f - uy);
                b = 0;
            }
            
            tempy = make_uchar4(255.0f * r, 255.0f * g, 255.0f * b, 255); // Use make_uchar4 for compact assignment
        }

        // Write outputs
        d_flowx_out[y * width + x] = tempx;
        d_flowy_out[y * width + x] = tempy;
    }
}