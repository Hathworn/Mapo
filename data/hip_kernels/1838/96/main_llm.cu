#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "normal_eqs_disparity_weighted_GPU_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *d_CD = NULL;
hipMalloc(&d_CD, XSIZE*YSIZE);
const float *d_disparity_compact = NULL;
hipMalloc(&d_disparity_compact, XSIZE*YSIZE);
const float4 *d_Zbuffer_normals_compact = NULL;
hipMalloc(&d_Zbuffer_normals_compact, XSIZE*YSIZE);
const int *d_ind_disparity_Zbuffer = NULL;
hipMalloc(&d_ind_disparity_Zbuffer, XSIZE*YSIZE);
float fx = 1;
float fy = 1;
float ox = 1;
float oy = 1;
float b = 2;
int n_cols = 1;
const int *d_n_values_disparity = NULL;
hipMalloc(&d_n_values_disparity, XSIZE*YSIZE);
const int *d_start_ind_disparity = NULL;
hipMalloc(&d_start_ind_disparity, XSIZE*YSIZE);
const float *d_abs_res_scales = NULL;
hipMalloc(&d_abs_res_scales, XSIZE*YSIZE);
float w_disp = 1;
const float *d_dTR = NULL;
hipMalloc(&d_dTR, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
normal_eqs_disparity_weighted_GPU<<<gridBlock, threadBlock>>>(d_CD,d_disparity_compact,d_Zbuffer_normals_compact,d_ind_disparity_Zbuffer,fx,fy,ox,oy,b,n_cols,d_n_values_disparity,d_start_ind_disparity,d_abs_res_scales,w_disp,d_dTR);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
normal_eqs_disparity_weighted_GPU<<<gridBlock, threadBlock>>>(d_CD,d_disparity_compact,d_Zbuffer_normals_compact,d_ind_disparity_Zbuffer,fx,fy,ox,oy,b,n_cols,d_n_values_disparity,d_start_ind_disparity,d_abs_res_scales,w_disp,d_dTR);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
normal_eqs_disparity_weighted_GPU<<<gridBlock, threadBlock>>>(d_CD,d_disparity_compact,d_Zbuffer_normals_compact,d_ind_disparity_Zbuffer,fx,fy,ox,oy,b,n_cols,d_n_values_disparity,d_start_ind_disparity,d_abs_res_scales,w_disp,d_dTR);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}