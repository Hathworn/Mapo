#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deInterleave_kernel(float *d_X_out, float *d_Y_out, float2 *d_XY_in, int pitch_out, int pitch_in, int width, int height) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) { // Check within image bounds using short-circuit AND
        float2 XY = *((float2*) ((char*) d_XY_in + y * pitch_in) + x); // Load input in a coalesced way
        float *out_x = (float*)((char*) d_X_out + y * pitch_out) + x;
        float *out_y = (float*)((char*) d_Y_out + y * pitch_out) + x;
        *out_x = XY.x; // Store X value
        *out_y = XY.y; // Store Y value
    }
}