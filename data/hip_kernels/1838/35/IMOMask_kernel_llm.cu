#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void IMOMask_kernel(float *d_IMOMask, float *d_IMO, const float *d_disparity, float offset, int n_cols, int n_rows) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < n_cols && y < n_rows) // Check if within image bounds
    {
        unsigned int ind = x + y * n_cols;
        if (d_IMOMask[ind] == 0.0f) { // Check if mask value is zero
            d_IMO[ind] = nanf(""); // Set NaN for invalid pixel
        }
    }
}