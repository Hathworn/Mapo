#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ac_kernel1 ( int *d_state_transition, unsigned int *d_state_supply, unsigned int *d_state_final, unsigned char *d_text, unsigned int *d_out, size_t pitch, int m, int n, int p_size, int alphabet, int numBlocks ) {

    // Calculate effective pitch
    int effective_pitch = pitch / sizeof(int);
    
    // Calculate characters per block and thread
    int charactersPerBlock = (n + numBlocks - 1) / numBlocks; // Use ceiling division
    int charactersPerThread = (charactersPerBlock + blockDim.x - 1) / blockDim.x; // Use ceiling division

    // Determine start and stop indices
    int startBlock = blockIdx.x * charactersPerBlock;
    int stopBlock = min(startBlock + charactersPerBlock, n); // Clamp to n
    int startThread = startBlock + charactersPerThread * threadIdx.x;
    int stopThread = min(startThread + charactersPerThread + m - 1, n); // Clamp to n

    int r = 0, s;
    int column;

    for ( column = startThread; column < stopThread; column++ ) {
        while ( ( s = d_state_transition[r * effective_pitch + (d_text[column] - (unsigned char)'A')] ) == -1 )
            r = d_state_supply[r];
        r = s;
        d_out[column] = d_state_final[r];
    }
}