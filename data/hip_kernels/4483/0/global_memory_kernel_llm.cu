#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_memory_kernel(int *d_go_to_state, unsigned int *d_failure_state, unsigned int *d_output_state, unsigned char *d_text, unsigned int *d_out, size_t pitch, int m, int n, int p_size, int alphabet, int num_blocks ) {

    // Calculate unique global thread index.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int effective_pitch = pitch / sizeof (int);

    int chars_per_block = n / num_blocks;

    int start_block = blockIdx.x * chars_per_block;
    int stop_block = min(n, start_block + chars_per_block); // Ensure bounds safety

    int chars_per_thread = (stop_block - start_block + blockDim.x - 1) / blockDim.x; // Ensure full coverage

    int start_thread = start_block + chars_per_thread * threadIdx.x;
    int stop_thread = min(n, start_thread + chars_per_thread + m - 1); // Ensure bounds safety

    int r = 0, s;

    for (int column = start_thread; (column < stop_thread && column < n); column++) {

        // Follow failure links until a valid goto state is found.
        while ((s = d_go_to_state[r * effective_pitch + (d_text[column] - (unsigned char)'A')]) == -1)
            r = d_failure_state[r];
        r = s;

        // Accumulate results for this thread.
        d_out[idx] += d_output_state[r];
    }
}