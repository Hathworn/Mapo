#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void encode(char *encodedText, char *decodedText) {
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Pre-calculate start positions and avoid recalculating inside the loop
    int startEncoded = id * 101;
    int startDecoded = id * 4;
    int finish = startEncoded + 100;

    // Bound check for valid thread ID
    if (id < 15360) {
        for (int t = startEncoded; t < finish; t++) {
            // Inline calculation, reducing unnecessary memory accesses
            if (encodedText[t] == ',') {
                decodedText[startDecoded++] = encodedText[t + 1];
            }
        }
    }
}