#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void mprts_update_offsets(int nr_total_blocks, uint *d_off, uint *d_spine_sums)
{
    // Calculate global thread index
    int bid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (bid < nr_total_blocks) {
        // Update offset for the current block
        d_off[bid] = d_spine_sums[bid * CUDA_BND_STRIDE];
    }
}