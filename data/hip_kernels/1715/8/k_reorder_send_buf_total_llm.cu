#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void k_reorder_send_buf_total(int nr_prts, int nr_total_blocks, uint *d_bidx, uint *d_sums, float4 *d_xi4, float4 *d_pxi4, float4 *d_xchg_xi4, float4 *d_xchg_pxi4)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified thread index calculation
    if (i >= nr_prts) return;  // Early exit if out of bounds

    if (d_bidx[i] == CUDA_BND_S_OOB) {
        int j = d_sums[i];  // Fetch the destination index
        d_xchg_xi4[j]  = d_xi4[i];  // Reorder xi4
        d_xchg_pxi4[j] = d_pxi4[i];  // Reorder pxi4
    }
}