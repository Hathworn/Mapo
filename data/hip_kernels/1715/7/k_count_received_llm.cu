#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void k_count_received(int nr_total_blocks, uint *d_n_recv_by_block, uint *d_spine_cnts)
{
    int bid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds before accessing array
    if (bid < nr_total_blocks) {
        int index = bid * 10 + CUDA_BND_S_NEW;
        d_spine_cnts[index] = d_n_recv_by_block[bid];
    }
}