#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void k_zero_comp_xyz(float *data, uint n, uint stride)
{
    // Using combined x and y dimensions for indexing
    uint idx = blockIdx.y * blockDim.x * gridDim.x + blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n * gridDim.y) {
        data[idx] = 0.f;
    }
}