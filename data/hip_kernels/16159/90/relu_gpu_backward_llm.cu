#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread ID is within bounds and perform computation
    if (tid < N) {
        float input_value = indata[tid];
        float output_gradient = outgrad[tid];
        ingrad[tid] = (input_value > 0) ? output_gradient : 0.0f;
    }
}