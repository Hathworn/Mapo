#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop through all elements assigned to this thread
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        out[i] = fmaxf(in[i], 0.0f); // Use fmaxf for better performance
    }
}