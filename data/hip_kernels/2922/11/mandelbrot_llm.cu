#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif
#ifdef __cplusplus
}
#endif
__global__ void mandelbrot(int* A, const int N, const int largeur, const int hauteur, const int start_hauteur, const int end_hauteur) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    int y = idx / largeur; // Correct the division for y
    int x = idx % largeur; // Use modulo for x to improve accuracy

    // Check bounds for both x and y
    if (y < (end_hauteur - start_hauteur) && x < largeur && idx < largeur * (end_hauteur - start_hauteur)) {
        int cpt = 0;
        float x1 = 0.0f;
        float y1 = 0.0f;
        
        // Pre-calculate scale factors for 'a' and 'b'
        float a = 4.0f * x / largeur - 2.0f;
        float b = 4.0f * (y + start_hauteur) / hauteur - 2.0f;

        float val = 0.0f; // Initialize val to 0.0f for better precision

        // Unroll loop partially for better performance
        #pragma unroll 4
        while (cpt < N && val <= 4.0f) {
            cpt++;
            float x2 = x1 * x1 - y1 * y1 + a;
            float y2 = 2.0f * x1 * y1 + b;
            x1 = x2;
            y1 = y2;
            val = x1* x1 + y1 * y1;
        }

        A[y * largeur + x] = cpt; // Correct index calculation
    }
}