#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_mult(const float* A, const float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N) // Ensure thread index is within array bounds
    {
        C[i] = A[i] * B[i]; // Perform element-wise multiplication
    }
}