#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

#ifdef __cplusplus
}
#endif

__global__ void vec_add_double(double *A, double *B, double* C, int size)
{
    // Calculate thread's unique global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure all threads are synchronized before performing vector addition
    if (index < size)
    {
        C[index] = A[index] + B[index];
    }
}