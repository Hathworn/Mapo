#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void spoc_max(const double* input, double* output, const int size)
{
    // Use shared memory to store intermediate maximums
    extern __shared__ double shared_data[];

    // Each thread loads one element into shared memory
    int tid = threadIdx.x;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size) {
        shared_data[tid] = fabs(input[i]);
    } else {
        shared_data[tid] = -INFINITY; // Handle out-of-bound threads
    }
    __syncthreads();

    // Perform reduction to find the maximum in each block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data[tid] = fmax(shared_data[tid], shared_data[tid + s]);
        }
        __syncthreads();
    }

    // Store block result in global memory by the first thread
    if (tid == 0) {
        output[blockIdx.x] = shared_data[0];
    }
}