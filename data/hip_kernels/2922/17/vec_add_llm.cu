#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif

#ifdef __cplusplus
}
#endif

__global__ void vec_add(float *A, float *B, float* C, int size)
{
    // Calculate the global thread index in a more readable way
    int index = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;

    // Use shared memory to optimize memory access
    extern __shared__ float sharedMem[];
    if (index < size) {
        sharedMem[threadIdx.x] = A[index] + B[index];
        __syncthreads();  // Ensure all threads have written to shared memory

        // Write the result to global memory
        C[index] = sharedMem[threadIdx.x];
    }
}