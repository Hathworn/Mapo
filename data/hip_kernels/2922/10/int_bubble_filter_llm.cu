#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void int_bubble_filter(int* input, const int* vec1, int* output, const int count)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int half_count = count / 2;
    if (tid < half_count)
    {
        output[tid * 2] = vec1[tid * 2];
        output[tid * 2 + 1] = vec1[tid * 2 + 1];
        __syncthreads(); // Ensure initial values are written before calculation begins

        for (int n = 0; n < 2 * count; n++)
        {
            int k = (n % 2 == 0) ? 0 : 1; // Replacing k = (k)?0:1; with n-based toggle
            int i = (tid * 2) + k;

            if (i + 1 < count)
            {
                if (!input[i] && input[i + 1])
                {
                    input[i] = 1;
                    input[i + 1] = 0;
                    output[i] = output[i + 1];
                    output[i + 1] = 0;
                }
                else
                {
                    if (!input[i])
                        output[i] = 0;
                    if (!input[i + 1])
                        output[i + 1] = 0;
                }
            }
            __syncthreads(); // Ensures all threads synchronize before next iteration
        }
    }
}