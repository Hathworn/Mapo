#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif
__global__ void sum(int* vec1, int* result, int* tmp1, const int count)
{
    // Calculate unique thread ID
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Calculate pair index for reduction
    int n = count / 2;

    if (tid < n) {
        // Initial reduction step
        tmp1[tid] = vec1[tid] + vec1[tid + n];
    }
    __syncthreads();

    // Loop for parallel reduction
    for (unsigned int stride = n / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            // Continuation of reduction using stride
            tmp1[tid] += tmp1[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        // Write result from thread 0
        *result = tmp1[0];
    }
}