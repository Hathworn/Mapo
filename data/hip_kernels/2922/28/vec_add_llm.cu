#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_add(const float* A, const float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Perform bounds checking before computation
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}