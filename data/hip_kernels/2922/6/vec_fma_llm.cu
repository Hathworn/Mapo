#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/





/****** Double precision *****/







#ifdef __cplusplus
}
#endif

__global__ void vec_fma(const float* A, const float* B, float* C, float* D, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        // Performing fused multiply-add operation
        D[i] = __fmaf_rn(B[i], C[i], A[i]);
    }
}