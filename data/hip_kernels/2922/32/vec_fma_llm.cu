#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/




/****** Double precision *****/



#ifdef __cplusplus
}
#endif

__global__ void vec_fma(const float* A, const float* B, float* C, float* D, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Add check to ensure boundary
    if (i >= N) return;

    // Combine addition and multiplication calculation
    D[i] = __fmul_rn(B[i], C[i]) + A[i];
}