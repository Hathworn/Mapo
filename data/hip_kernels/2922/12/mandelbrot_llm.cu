#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif



#ifdef __cplusplus
}
#endif
__global__ void mandelbrot(int* A, const int N, const int largeur, const int hauteur){
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= largeur * hauteur) return; // Early exit if index is out of bounds

    int x = idx % largeur;   // Use modulo for x
    int y = idx / largeur;   // Calculate y based on index

    int cpt = 0;
    float x1 = 0.0f;
    float y1 = 0.0f;
    float x2 = 0.0f;
    float y2 = 0.0f;
    float a = 4.0f * x / largeur - 2.0f;
    float b = 4.0f * y / hauteur - 2.0f;

    float val = x1 * x1 + y1 * y1;

    while (cpt < N && val <= 4.0f) {
        cpt++;
        x2 = x1 * x1 - y1 * y1 + a;
        y2 = 2.0f * x1 * y1 + b;
        x1 = x2;
        y1 = y2;
        val = x1 * x1 + y1 * y1;
    }

    A[y * largeur + x] = cpt; // Correct index for row-major order storage
}