#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

#ifdef __cplusplus
}
#endif

__global__ void vec_sub(const float* A, const float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Ensure computation only for valid index
    if (i < N) {
        C[i] = A[i] - B[i];
    }
}