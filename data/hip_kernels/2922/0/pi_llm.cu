#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif

struct point{
float x;
float y;
};

struct point2{
double x;
double y;
};

#ifdef __cplusplus
}
#endif
__global__ void pi(const struct point* A, int* res, const int nbPoint, const float ray) {
    const int idx_start = 32 * blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * 32;
    int idx = idx_start;

    // Loop unrolling for performance
    #pragma unroll 16
    for (int j = 0; j < 32 && idx < nbPoint; j++, idx += blockDim.x) {
        float ax = A[idx].x;
        float ay = A[idx].y;
        res[idx] = (ax * ax + ay * ay <= ray);
    }
}