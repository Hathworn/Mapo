#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_sub(const float* A, const float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Check if the index is within bounds and perform subtraction
    if (i < N) {
        C[i] = A[i] - B[i];  // Direct memory access for subtraction
    }
}