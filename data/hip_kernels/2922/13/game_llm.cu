#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif



#ifdef __cplusplus
}
#endif
__global__ void game(int* A, const int N, const int largeur, const int hauteur) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Precalculation to improve readability and prevent repetitive operations
    int y = idx / largeur;
    int x = idx % largeur;

    if (y >= hauteur || x >= largeur)
        return;

    int me = A[idx];
    int north = (y > 0) ? A[idx - largeur] : 0;
    int south = (y < hauteur - 1) ? A[idx + largeur] : 0;
    int west = (x > 0) ? A[idx - 1] : 0;
    int east = (x < largeur - 1) ? A[idx + 1] : 0;

    // Calculate diagonal neighbors with boundary checks
    int northEast = (y > 0 && x < largeur - 1) ? A[idx - largeur + 1] : 0;
    int northWest = (y > 0 && x > 0) ? A[idx - largeur - 1] : 0;
    int southEast = (y < hauteur - 1 && x < largeur - 1) ? A[idx + largeur + 1] : 0;
    int southWest = (y < hauteur - 1 && x > 0) ? A[idx + largeur - 1] : 0;

    // Calculate the sum of neighbor states
    int res = north + south + east + west + northEast + northWest + southEast + southWest;

    // Game of Life rules
    if ((me == 1 && (res < 2 || res > 3)) || (me == 0 && res == 3))
        A[idx] = !me;
}