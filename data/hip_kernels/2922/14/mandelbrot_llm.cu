#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif
#ifdef __cplusplus
}
#endif

// Optimize memory access by using shared memory for largeur and hauteur.
__global__ void mandelbrot(int* A, const int N, const int largeur, const int hauteur) {
    __shared__ int shared_largeur, shared_hauteur;
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        shared_largeur = largeur;
        shared_hauteur = hauteur;
    }
    __syncthreads();

    int y = threadIdx.y + (blockIdx.y * blockDim.y);
    int x = threadIdx.x + (blockIdx.x * blockDim.x);

    if (y < shared_hauteur && x < shared_largeur) {
        int cpt = 0;
        float x1 = 0.f;
        float y1 = 0.f;
        float x2 = 0.f;
        float y2 = 0.f;

        // Precompute constants for efficiency
        float scaled_x = 4.f / shared_largeur * x - 2.f;
        float scaled_y = 4.f / shared_hauteur * y - 2.f;

        float val = x1 * x1 + y1 * y1;

        while (cpt < N && val <= 4.f) {
            cpt++;
            x2 = x1 * x1 - y1 * y1 + scaled_x;
            y2 = 2.f * x1 * y1 + scaled_y;
            x1 = x2;
            y1 = y2;
            val = x1 * x1 + y1 * y1;
        }

        A[y * shared_hauteur + x] = cpt;
    }
}