#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_div(const float* A, const float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        // Optimize with conditional statement removed from the loop body
        C[i] = A[i] / B[i];
    }
}