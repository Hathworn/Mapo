#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

#ifdef __cplusplus
}
#endif

__global__ void int_bubble_filter(int* input, const int* vec1, int* output, const int count)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < count / 2) // Use '<' instead of '<=' for proper range coverage
    {
        int base = tid * 2;
        output[base] = vec1[base];
        output[base + 1] = vec1[base + 1];

        for (int n = 0; n < count * 2; n++)
        {
            int i = base + (n & 1); // Use bitwise operation for k toggle
            if (i + 1 < count)
            {
                int val1 = input[i];
                int val2 = input[i + 1];

                if ((!val1) && (val2))
                {
                    input[i] = 1;
                    input[i + 1] = 0;
                    output[i] = output[i + 1];
                    output[i + 1] = 0;
                }
                else
                {
                    if (!val1) output[i] = 0;
                    if (!val2) output[i + 1] = 0;
                }
            }
            __syncthreads();
        }
    }
}