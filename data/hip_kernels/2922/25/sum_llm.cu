#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif
__global__ void sum(int* vec1, int* result, int* tmp1, const int count) 
{
    // Calculate global thread ID
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Load elements into shared memory for faster access
    extern __shared__ int shared[];
    int n = count / 2;
    if (tid < n)
        shared[tid] = vec1[tid] + vec1[tid + n];
    else
        shared[tid] = 0;
    __syncthreads();

    // Perform parallel reduction using shared memory
    for (unsigned int stride = n / 2; stride > 0; stride >>= 1) {
        if (tid < stride)
            shared[tid] += shared[tid + stride];
        __syncthreads();
    }
    
    // Store result back to global memory
    if (tid == 0)
        *result = shared[0];
}