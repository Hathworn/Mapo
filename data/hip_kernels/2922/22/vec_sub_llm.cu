#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/





/****** Double precision *****/







#ifdef __cplusplus
}
#endif
__global__ void vec_sub(const float* A, const float* B, float* C, int N) {
    // Calculate global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Optimize condition check by placing it inside the if block
    if (i < N) {
        C[i] = A[i] - B[i];
    }
}