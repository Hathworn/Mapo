#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

#ifdef __cplusplus
}
#endif

__global__ void sum(int *vec1, int *result, int *tmp1, const int count)
{
    // Calculate global thread ID
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Perform first step of parallel reduction
    if (tid < count / 2) {
        tmp1[tid] = vec1[tid] + vec1[tid + count / 2];
    }
    __syncthreads();

    // Perform the reduction using shared memory if possible
    for (unsigned int stride = count / 4; stride > 0; stride /= 2) {
        if (tid < stride) {
            tmp1[tid] += tmp1[tid + stride];
        }
        __syncthreads();
    }

    // Write the final result
    if (tid == 0) {
        *result = tmp1[0];
    }
}