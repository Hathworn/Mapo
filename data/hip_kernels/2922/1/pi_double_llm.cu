#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif

struct point{
    float x;
    float y;
};

struct point2{
    double x;
    double y;
};

#ifdef __cplusplus
}
#endif

__global__ void pi_double(const struct point2* A, int* res, const int nbPoint, const float ray) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;  // Simplified index calculation
    const double raySquared = (double)ray * ray;  // Precompute squared radius
    const int stride = 32 * blockDim.x;  // Define stride

    for (int globalIdx = idx; globalIdx < nbPoint; globalIdx += stride) {  // Global thread stride loop
        int localIdx = globalIdx;
        if (localIdx < nbPoint) {
            res[localIdx] = (A[localIdx].x * A[localIdx].x + A[localIdx].y * A[localIdx].y <= raySquared);
        }
    }
}