#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/





/****** Double precision *****/






#ifdef __cplusplus
}
#endif

__global__ void vec_add_64(const double* A, const double* B, double* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure thread does useful work within bounds
    if (i < N) {
        // Perform efficient memory access and vector addition
        C[i] = A[i] + B[i];
    }
}