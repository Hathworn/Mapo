#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

#ifdef __cplusplus
}
#endif

__global__ void init(double* A, double* V, double* VN, int size, int n) {
    // Calculate the global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
   
    // Check bounds
    if (i >= size) return;
    
    // Use a register for accumulation for better performance
    double s = 0.0;
    
    // Iterate through 'n' and accumulate in 's'
    for (int j = 0; j < n; j++) {
        s += A[i * n + j] * V[j];
    }
    
    // Write the result back
    VN[i] = s;
}