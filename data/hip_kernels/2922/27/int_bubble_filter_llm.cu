#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/





/****** Double precision *****/







#ifdef __cplusplus
}
#endif
__global__ void int_bubble_filter(int* input, const int* vec1, int* output, const int count)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimize by unrolling the loop and using shared memory
    if (tid < count / 2)
    {
        output[tid * 2] = vec1[tid * 2];
        output[tid * 2 + 1] = vec1[tid * 2 + 1];

        extern __shared__ int s_input[];  // Shared memory for input array
        extern __shared__ int s_output[]; // Shared memory for output array

        s_input[tid * 2] = input[tid * 2];
        s_input[tid * 2 + 1] = input[tid * 2 + 1];
        s_output[tid * 2] = output[tid * 2];
        s_output[tid * 2 + 1] = output[tid * 2 + 1];

        __syncthreads();

        for (int n = 0; n < count*2; n++)
        {
            int k = (n % 2);
            int i = (tid * 2) + k;
            if (i + 1 < count)
            {
                if ((!s_input[i]) && (s_input[i + 1]))
                {
                    s_input[i] = 1;
                    s_input[i + 1] = 0;
                    s_output[i] = s_output[i + 1];
                    s_output[i + 1] = 0;
                }
                else
                {
                    if (!s_input[i])
                        s_output[i] = 0;
                    if (!s_input[i + 1])
                        s_output[i + 1] = 0;
                }
            }
            __syncthreads();
        }

        output[tid * 2] = s_output[tid * 2];
        output[tid * 2 + 1] = s_output[tid * 2 + 1];
    }
}