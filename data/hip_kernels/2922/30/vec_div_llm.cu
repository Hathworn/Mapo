#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_div(const float* A, const float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        // Optimize division by checking for zero in B
        float b_val = B[i];
        C[i] = (b_val != 0.0f) ? (A[i] / b_val) : 0.0f; // Avoid division by zero
    }
}