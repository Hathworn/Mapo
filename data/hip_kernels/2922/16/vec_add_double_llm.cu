#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif

#ifdef __cplusplus
}
#endif

__global__ void vec_add_double(double *A, double *B, double* C, int size)
{
    // Calculate global thread ID
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop for efficiency, process more elements per thread
    for (int i = index; i < size; i += stride) {
        C[i] = A[i] + B[i];
    }
}