#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __cplusplus
extern "C" {
#endif

#ifdef __cplusplus
}
#endif

__global__ void vec_add(float *A, float *B, float* C, int size)
{
    // Calculate global index using grid stride loop
    for (int index = blockIdx.x * blockDim.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x)
    {
        C[index] = A[index] + B[index];
    }
}