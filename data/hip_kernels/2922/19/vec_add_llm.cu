#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_add(const float* A, const float* B, float* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure the thread index is within bounds
    if (i < N) {
        // Perform vector addition
        C[i] = A[i] + B[i];
    }
}