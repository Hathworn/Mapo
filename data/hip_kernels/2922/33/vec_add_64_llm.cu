#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/





/****** Double precision *****/







#ifdef __cplusplus
}
#endif

__global__ void vec_add_64(const double* A, const double* B, double* C, int N) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        // Perform addition when the index is within bounds
        C[i] = A[i] + B[i];
    }
}