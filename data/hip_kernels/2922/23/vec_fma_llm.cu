#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

/****** Single precision *****/

/****** Double precision *****/

#ifdef __cplusplus
}
#endif

__global__ void vec_fma(const float* A, const float* B, float* C, float* D, int N) {
    // Calculate global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Thread index is within bounds
    if (i < N) {
        // Perform fused multiply-add operation
        D[i] = A[i] + B[i] * C[i];
    }
}