#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

#ifdef __cplusplus
}
#endif

__global__ void spoc_max(const double* input, double* output, const int size)
{
    // Use shared memory for intermediate results
    extern __shared__ double sdata[];

    int tid = threadIdx.x;
    int i = blockDim.x * blockIdx.x + tid;
    
    // Initialize shared memory
    sdata[tid] = (i < size) ? fabs(input[i]) : 0.0;
    __syncthreads();

    // Reduce within block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = fmax(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) output[blockIdx.x] = sdata[0];
}