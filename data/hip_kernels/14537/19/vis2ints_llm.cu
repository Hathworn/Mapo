#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vis2ints(double scale, double2 *vis_in, int2* vis_out, int npts) {
    // Cache thread and block indices
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int stride = gridDim.x * blockDim.x;
    
    for (int q = tid; q < npts; q += stride) {
        double2 inn = vis_in[q];
        
        // Scale inputs once and use registers effectively
        double scaled_x = inn.x * scale;
        double scaled_y = inn.y * scale;
        
        int main_y = __double2int_rd(scaled_y);                 // Use fast rounding
        int sub_y = __double2int_rd(GCF_GRID * (scaled_y - main_y));
        int main_x = __double2int_rd(scaled_x);
        int sub_x = __double2int_rd(GCF_GRID * (scaled_x - main_x));
        
        // Directly assign to output using calculated indices
        vis_out[q].x = main_x * GCF_GRID + sub_x;
        vis_out[q].y = main_y * GCF_GRID + sub_y;
    }
}