#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_bookmarks(int2* vis_in, int npts, int blocksize, int blockgrid, int* bookmarks) {
    // Cache commonly used values
    int grid_stride = gridDim.x * blockDim.x;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Use shared memory to reduce global memory access
    __shared__ int shared_last_vis_x, shared_last_vis_y;

    for (int q = idx; q <= npts; q += grid_stride) {
        int2 this_vis = vis_in[q];

        // Use a thread to initialize shared memory for the first element
        if (q == 0) {
            shared_last_vis_x = -1;
            shared_last_vis_y = 0;
        } else {
            shared_last_vis_x = vis_in[q-1].x / GCF_GRID / blocksize;
            shared_last_vis_y = vis_in[q-1].y / GCF_GRID / blocksize;
        }
        __syncthreads(); // Make sure shared_last_vis_x and shared_last_vis_y are set

        int main_x = this_vis.x / GCF_GRID / blocksize;
        int main_y = this_vis.y / GCF_GRID / blocksize;

        if (npts == q) {
            main_x = main_y = blockgrid;
        }

        if (main_x != shared_last_vis_x || main_y != shared_last_vis_y) {
            for (int z = shared_last_vis_y * blockgrid + shared_last_vis_x + 1; 
                 z <= main_y * blockgrid + main_x; z++) {
                bookmarks[z] = q;
            }
        }
    }
}