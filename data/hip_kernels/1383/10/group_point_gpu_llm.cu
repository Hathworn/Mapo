#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_idx = blockIdx.x; // Assign batch index to each block
    int sample_idx = threadIdx.x; // Assign sample index to each thread

    if (batch_idx < b && sample_idx < nsample * m) {
        int j = sample_idx / nsample; // Calculate m index
        int k = sample_idx % nsample; // Calculate nsample index

        int ii = idx[j * nsample + k]; // Get index from idx array

        for (int l = 0; l < c; ++l) { // Unroll innermost loop for better ILP
            out[j * nsample * c + k * c + l] = points[ii * c + l];
        }
    }
}