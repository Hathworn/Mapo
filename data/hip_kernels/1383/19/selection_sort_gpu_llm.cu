#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, float *dist, int *idx, float *val) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    idx += m * k * batch_index;
    val += m * k * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int j = index; j < m; j += stride) {
        float *p_dist = dist + j * n;

        // Optimized selection sort for the first k elements
        for (int s = 0; s < k; ++s) {
            int min = s;
            float min_val = p_dist[s];

            // Find the min
            for (int t = s + 1; t < n; ++t) {
                if (p_dist[t] < min_val) {
                    min = t;
                    min_val = p_dist[t];
                }
            }

            // Update idx and val
            idx[j * k + s] = min; // Adjusted to stride by k
            val[j * k + s] = min_val;

            // Swap min-th and s-th element
            p_dist[min] = p_dist[s];
            p_dist[s] = min_val; // Use min_val directly
        }
    }
}