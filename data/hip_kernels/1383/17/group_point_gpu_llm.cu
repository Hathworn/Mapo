#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batchIndex = blockIdx.x;
    int threadIndex = threadIdx.x;

    points += n * c * batchIndex;
    idx += m * nsample * batchIndex;
    out += m * nsample * c * batchIndex;
    
    // Assigning each thread to handle a single sample from m * nsample
    int globalIndex = threadIndex;
    while (globalIndex < m * nsample) {
        int j = globalIndex / nsample;
        int k = globalIndex % nsample;
        int ii = idx[j * nsample + k];
        for (int l = 0; l < c; ++l) {
            out[j * nsample * c + k * c + l] = points[ii * c + l];
        }
        globalIndex += blockDim.x;
    }
}