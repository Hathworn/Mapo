#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    outi += m * n * batch_index;
    out += m * n * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Coalesced memory access for copying
    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            float dist_value = dist[j * n + s];
            out[j * n + s] = dist_value;
            outi[j * n + s] = s;
        }
    }

    // Reduce divergent branches by reordering loops
    for (int j = index; j < m; j += stride) {
        float *p_dist = out + j * n;
        for (int s = 0; s < k; ++s) {
            int min = s;
            float minValue = p_dist[s];
            for (int t = s + 1; t < n; ++t) {
                float tmpValue = p_dist[t];
                if (tmpValue < minValue) {
                    min = t;
                    minValue = tmpValue;
                }
            }
            // Swap only if needed
            if (min != s) {
                p_dist[min] = p_dist[s];
                p_dist[s] = minValue;
                int tmpi = outi[j * n + min];
                outi[j * n + min] = outi[j * n + s];
                outi[j * n + s] = tmpi;
            }
        }
    }
}