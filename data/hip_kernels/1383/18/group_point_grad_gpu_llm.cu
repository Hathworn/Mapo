#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    // Calculate global thread index
    int batch_index = blockIdx.x;
    int thread_index = threadIdx.x;

    // Adjust pointers to this batch
    idx += batch_index * m * nsample;
    grad_out += batch_index * m * nsample * c;
    grad_points += batch_index * n * c;

    // Use shared memory to optimize access (assumed cumulative initialization outside kernel)
    __shared__ float shared_grad_out[1024];  // Size depending on actual use case
    __shared__ int shared_idx[1024];         // Size depending on actual use case

    for (int j = thread_index; j < m; j += blockDim.x) {
        for (int k = 0; k < nsample; ++k) {
            int point_idx = shared_idx[j * nsample + k];
            for (int l = 0; l < c; ++l) {
                // Optimize by using shared memory and coalesced access
                atomicAdd(&grad_points[point_idx * c + l], shared_grad_out[j * nsample * c + k * c + l]);
            }
        }
    }
}