#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_idx = blockIdx.x; // Each block handles one batch
    int j = blockIdx.y; // Each thread block column handles one output column (m)
    int k = threadIdx.x; // Each thread handles one sample in nsample

    if (batch_idx < b && j < m && k < nsample) {
        int ii = idx[j * nsample + k];
        for (int l = 0; l < c; ++l) {
            atomicAdd(&grad_points[ii * c + l], grad_out[j * nsample * c + k * c + l]);
        }
    }
}
```
