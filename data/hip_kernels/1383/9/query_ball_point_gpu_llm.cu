#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    // Calculate thread index
    int batch_index = blockIdx.x;
    int point_index = blockIdx.y * blockDim.x + threadIdx.x;

    if (batch_index < b && point_index < m) {
        int cnt = 0;
        float x2 = xyz2[point_index * 3 + 0];
        float y2 = xyz2[point_index * 3 + 1];
        float z2 = xyz2[point_index * 3 + 2];

        for (int k = 0; k < n && cnt < nsample; ++k) {
            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            float d = sqrtf((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1));

            if (d < radius) {
                if (cnt == 0) { 
                    // Duplicate index initialization for valid cases
                    for (int l = 0; l < nsample; ++l)
                        idx[point_index * nsample + l] = k;
                }
                idx[point_index * nsample + cnt] = k;
                cnt++;
            }
        }
    }
}