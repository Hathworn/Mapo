#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    // Calculate batch offset
    int batch_index = blockIdx.x;
    xyz1 += n * 3 * batch_index;
    xyz2 += m * 3 * batch_index;
    idx += m * nsample * batch_index;

    // Parallelizing over m
    int index = threadIdx.x + blockIdx.y * blockDim.x; // Use 2D grid for better occupancy
    int stride = blockDim.x * gridDim.y; // Update stride based on 2D grid
    
    for (int j = index; j < m; j += stride) {
        int cnt = 0;
        float x2 = xyz2[j * 3 + 0];
        float y2 = xyz2[j * 3 + 1];
        float z2 = xyz2[j * 3 + 2];
        
        // Unroll loop partially to reduce branching
        for (int k = 0; k < n; ++k) {
            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            float dx = x2 - x1;
            float dy = y2 - y1;
            float dz = z2 - z1;
            float d2 = dx * dx + dy * dy + dz * dz; // Avoid sqrt for distance comparison

            if (d2 < radius * radius) { // Compare squared radius
                if (cnt == 0) {
                    // Set all indices to k initially
                    for (int l = 0; l < nsample; ++l) {
                        idx[j * nsample + l] = k;
                    }
                }
                idx[j * nsample + cnt] = k;
                cnt++;

                if (cnt == nsample) break; // Break if nsample points are found
            }
        }
    }
}