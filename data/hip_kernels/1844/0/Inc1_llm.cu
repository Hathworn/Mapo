#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc1(float *Ad, float *Bd) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx < 1) {
        float tempA = Ad[tx]; // Load Ad[tx] once
        for (int i = 0; i < ITER; i++) {
            tempA += 1.0f; // Increment tempA instead of Ad[tx]
            __syncthreads(); // Ensure all threads have synchronized here
            for (int j = 0; j < 256; j++) {
                Bd[tx] = tempA; // Update Bd[tx] once in each iteration
            }
            __syncthreads(); // Synchronize after the Bd[tx] update
        }
        Ad[tx] = tempA; // Store final result back to Ad[tx]
    }
}