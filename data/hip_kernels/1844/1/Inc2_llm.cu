#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc2(float *Ad, float *Bd) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx < 1024) {
        // Use register to store value and reduce global memory access
        float a = Ad[tx];
        for (int i = 0; i < ITER; i++) {
            a += 1.0f; // Accumulate in register
            // Only update Bd after inner loop to reduce memory write
            for (int j = 0; j < 256; j++) {
                Bd[tx] = a;
            }
        }
        Ad[tx] = a; // Write back final result to Ad
    }
}