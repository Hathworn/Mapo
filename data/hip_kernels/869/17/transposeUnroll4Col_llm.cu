#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Col(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = (blockDim.x * blockIdx.x + threadIdx.x) * 4; // Calculate ix with unrolled factor
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    if (ix + 3 < nx && iy < ny) // Ensure ix and iy are within bounds
    {
        // Efficient memory fetch and store using unrolling
        float4 data;
        data.x = in[to];
        data.y = in[to + ny];
        data.z = in[to + 2 * ny];
        data.w = in[to + 3 * ny];

        out[ti] = data.x;
        out[ti + blockDim.x] = data.y;
        out[ti + 2 * blockDim.x] = data.z;
        out[ti + 3 * blockDim.x] = data.w;
    }
}