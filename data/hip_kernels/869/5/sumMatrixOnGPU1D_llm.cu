#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure thread accesses within matrix boundary
    if (ix < nx) {
        // Use a stride to iterate over rows handled by the same thread
        for (int iy = 0; iy < ny; iy++) {
            int idx = iy * nx + ix;
            MatC[idx] = MatA[idx] + MatB[idx];
        }
    }
}