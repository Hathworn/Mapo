#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Cache blockDim.x in a local variable for reuse
    unsigned int blockDimX = blockDim.x;

    // Calculate the global index for the x direction
    unsigned int ix = threadIdx.x + blockIdx.x * blockDimX;

    // Calculate the global index for the y direction
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate the flattened index
    unsigned int idx = iy * nx + ix;

    // Check boundary conditions to avoid accessing invalid memory
    if (ix < nx && iy < ny)
    {
        // Perform element-wise addition
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}