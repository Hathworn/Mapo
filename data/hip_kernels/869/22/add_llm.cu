#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *c, int *d) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Use blockIdx to calculate global thread ID

    if (tid < N) { // Add bounds check for safety
        d[tid] += c[tid];
    }
}
```
Note: Ensure `N` is properly defined to represent the total number of elements.