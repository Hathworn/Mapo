#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate the unique thread index in a 2D grid
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    // Use a thread block that ensures all boundary checks are included
    if (ix < nx && iy < ny) {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}