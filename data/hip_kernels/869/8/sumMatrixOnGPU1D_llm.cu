#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;

    // Use stride to handle whole matrix efficiently
    for (int idx = ix; idx < nx * ny; idx += blockDim.x * gridDim.x)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}