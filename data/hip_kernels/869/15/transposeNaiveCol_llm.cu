#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNaiveCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate the thread's unique global index.
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure indices are within matrix bounds.
    if (ix < nx && iy < ny)
    {
        // Perform matrix transposition.
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}