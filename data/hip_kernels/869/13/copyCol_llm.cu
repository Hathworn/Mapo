#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate flattened thread index for improved memory access pattern
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int offset = blockDim.x * gridDim.x;

    // Iterate over columns with stride to ensure all elements are copied
    while (idx < nx * ny)
    {
        out[idx] = in[idx];
        idx += offset; // Move to next column position
    }
}