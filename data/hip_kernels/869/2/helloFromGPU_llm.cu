#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU()
{
    // Use threadIdx and blockIdx for potentially unique messages
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello World from GPU! My index is: %d\n", index);
}