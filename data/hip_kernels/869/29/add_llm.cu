#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add (float *d_A, float *d_B, float *d_C, int widthA, int widthB, int widthC)
{
    // Optimize memory access by using cooperative fetching and vectorized loads/stores
    int globalIndex = blockIdx.x * 64 + threadIdx.x * 2 + (blockIdx.y * 8 + threadIdx.y) * widthA;
    
    float2 tempA = *(reinterpret_cast<float2*>(d_A + globalIndex));
    float2 tempB = *(reinterpret_cast<float2*>(d_B + globalIndex));
    
    tempA.x += tempB.x;
    tempA.y += tempB.y;

    *(reinterpret_cast<float2*>(d_C + globalIndex)) = tempA;
}