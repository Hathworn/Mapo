#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyRow(float *out, float *in, const int nx, const int ny)
{
    // Calculate single index for multidimensional array
    unsigned int idx = blockIdx.y * blockDim.y * nx + threadIdx.y * nx + blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single index and boundary check once
    if (idx < nx * ny)
    {
        out[idx] = in[idx];
    }
}