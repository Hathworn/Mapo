#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate each thread's data point in the matrix more efficiently
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix < nx && iy < ny) {
        unsigned int idx = iy * nx + ix; // Compute index only if within bounds
        MatC[idx] = MatA[idx] + MatB[idx]; // Directly perform addition
    }
}