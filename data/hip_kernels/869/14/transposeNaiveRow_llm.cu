#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaiveRow(float *out, float *in, const int nx, const int ny)
{
    // Cache less frequently accessed dimensions to registers
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Check bounds and assign
    if (ix < nx && iy < ny)
    {
        // Reorder memory access for coalescing
        unsigned int inputIndex = iy * nx + ix;
        unsigned int outputIndex = ix * ny + iy;
        out[outputIndex] = in[inputIndex];
    }
}