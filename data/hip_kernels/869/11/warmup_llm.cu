#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(float *out, float *in, const int nx, const int ny)
{
    // Calculate the unique thread index
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix; // Flatten 2D index to 1D

    // Perform operation only if within array bounds
    if (ix < nx && iy < ny)
    {
        out[idx] = in[idx];
    }
}