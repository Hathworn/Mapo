#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized transposeDiagonalCol kernel function
__global__ void transposeDiagonalCol(float *out, float *in, const int nx, const int ny)
{
    unsigned int blk_x = blockIdx.x;
    unsigned int blk_y = blockIdx.y;

    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    // Combined block and thread indices for better access pattern
    if (ix < nx && iy < ny)
    {
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}