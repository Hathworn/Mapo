#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // Utilize shared memory for faster access
    extern __shared__ float sharedMemory[];

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory to minimize repeated global memory access
    if (i < N) {
        sharedMemory[threadIdx.x] = A[i];
        __syncthreads();
        C[i] = sharedMemory[threadIdx.x] + B[i];
    }
}