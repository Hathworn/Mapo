#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalRow(float *out, float *in, const int nx, const int ny)
{
    // Cache block index calculations
    unsigned int blkIdxX = blockIdx.x;
    unsigned int blkIdxY = blockIdx.y;
    unsigned int blk_x = (blkIdxX + blkIdxY) % gridDim.x;

    // Calculate global thread index
    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blkIdxX + threadIdx.y; // Correct block index for y

    // Check bounds and perform transpose
    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}