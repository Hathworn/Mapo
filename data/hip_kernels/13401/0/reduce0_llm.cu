#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int N = 256;
const int C = 1024;
const int D = 28*28;

__global__ void reduce0(const float* in, float* out) {
    __shared__ float buffer[CUDA_NUM_THREADS];
    const unsigned int tid = threadIdx.x;
    const unsigned int c = blockIdx.x;

    // Initialize shared memory buffer
    buffer[tid] = 0.0f;

    // Efficient data loading to buffer using strided access pattern
    for (int i = tid; i < N * D; i += blockDim.x) {
        const unsigned int n = i / D;
        const unsigned int d = i % D;
        const unsigned int index = n * C * D + c * D + d;
        buffer[tid] += in[index];
    }
    __syncthreads();

    // Tree-based reduction for buffer with non-divergent branching
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            buffer[tid] += buffer[tid + s];
        }
        __syncthreads();
    }

    // Write result to out for the first thread
    if (tid == 0) out[c] = buffer[0] / (N * D);
}