#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int N = 256;
const int C = 1024;
const int D = 28*28;

__global__ void reduce1(const float* in, float* out) {
    extern __shared__ float buffer[];
    const unsigned int tid = threadIdx.x;
    const unsigned int c = blockIdx.x;
    float sum = 0.0f;

    // Load and accumulate data to buffer
    for (int i = tid; i < N * D; i += blockDim.x) {
        const unsigned int n = i / D;
        const unsigned int d = i % D;
        const unsigned int index = n * C * D + c * D + d;
        sum += in[index];
    }
    buffer[tid] = sum;
    __syncthreads();

    // Optimized tree reduction with stride halving
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {  // Reduce bank conflicts with stride halving
            buffer[tid] += buffer[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) out[c] = buffer[0] / (N * D);
}