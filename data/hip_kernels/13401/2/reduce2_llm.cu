#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int N = 256;
const int C = 1024;
const int D = 28*28;

__global__ void reduce2(const float* __restrict__ in, float* __restrict__ out) {
    __shared__ float buffer[CUDA_NUM_THREADS];
    const unsigned int tid = threadIdx.x;
    const unsigned int c = blockIdx.x;

    // load and accumulate data to buffer
    float sum = 0.0f;
    for (int i = tid; i < N * D; i += blockDim.x) {
        const unsigned int n = i / D;
        const unsigned int d = i % D;
        const unsigned int index = n * C * D + c * D + d;
        sum += in[index];
    }
    buffer[tid] = sum;
    __syncthreads();

    // perform inter-thread reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            buffer[tid] += buffer[tid + s];
        }
        __syncthreads();
    }

    // write the result to global memory
    if (tid == 0) 
        out[c] = buffer[0] / (N * D);
}