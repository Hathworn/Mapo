#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod, const size_t lod_len, size_t* out_lod0) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is within array bounds and access memory efficiently
    if (index < lod_len) {
        size_t in_lod_value = in_lod[index];  // Cache in_lod value
        out_lod0[index] = in_lod_value - num_erased[in_lod_value];
    }
}