#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid-stride loop for better occupancy and resource utilization
    for (; tid < N; tid += gridDim.x * blockDim.x) {
        out[tid] = in[tid] > 0 ? in[tid] : 0;
    }
}