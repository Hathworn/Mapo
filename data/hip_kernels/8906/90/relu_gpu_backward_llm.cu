#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use cooperative thread group stride to cover all elements
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        ingrad[i] = indata[i] > 0 ? outgrad[i] : 0;
    }
}