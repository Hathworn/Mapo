#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
*
*
* This code is based on the NVIDIA 'reduction' CUDA sample,
* Copyright 1993-2010 NVIDIA Corporation.
*/

extern "C"

__global__ void setTargetIndexNormalize(int n, double sum, double *w, double *out, double* output)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for better memory access efficiency
    __shared__ double out_shared[256];
    
    if (i < n)
    {
        // Load data into shared memory
        out_shared[threadIdx.x] = w[i] == 1.0 ? out[i] : 0.0;
        __syncthreads(); // Synchronize to ensure all threads have loaded their values
        
        // First thread in block computes the result
        if(threadIdx.x == 0) 
        {
            double tempSum = 0.0;
            for(int j = 0; j < blockDim.x; j++)
            {
                if(i + j < n) // Check for boundary within block
                {
                    tempSum += out_shared[j];
                }
            }
            // Update output[0] atomically to avoid race conditions
            atomicAdd(&output[0], tempSum / sum);
        }
    }
}