#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
*
* This code is based on the NVIDIA 'reduction' CUDA sample,
* Copyright 1993-2010 NVIDIA Corporation.
*/

extern "C"

__global__ void getTargetIndex(int n, int *index, double *w)
{
    // Use shared memory for index to reduce global memory access
    __shared__ int sharedIndex;

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        if (w[i] == 1.0) 
        {
            sharedIndex = i;
        }
    }

    __syncthreads();

    // Use atomic operation to safely update index in global memory
    if (threadIdx.x == 0 && sharedIndex != -1)
    {
        atomicExch(&index[0], sharedIndex);
    }
}