#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* 
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
* 
* 
* This code is based on the NVIDIA 'reduction' CUDA sample,
* Copyright 1993-2010 NVIDIA Corporation.
*/

extern "C"

__global__ void difference(int n, double *actual, double *target, double* out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if the index is within bounds
    if (i < n)
    {
        // Directly write result to output array
        out[i] = actual[i] - target[i];
    }
}