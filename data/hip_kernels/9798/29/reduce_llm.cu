#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
*
*
* This code is based on the NVIDIA 'reduction' CUDA sample,
* Copyright 1993-2010 NVIDIA Corporation.
*/

extern "C"
__global__ void reduce(double *g_idata, double *g_odata, unsigned int n)
{
    extern __shared__ double sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    unsigned int gridSize = blockDim.x * 2 * gridDim.x;

    double mySum = 0;

    // Loop optimization: Unroll loop for better performance
    while (i < n)
    {
        mySum += g_idata[i];
        if (i + blockDim.x < n) mySum += g_idata[i + blockDim.x];
        i += gridSize;
    }

    // Write to shared memory
    sdata[tid] = mySum;
    __syncthreads();

    // Perform reduction in shared memory
    if (blockDim.x >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockDim.x >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockDim.x >= 128) { if (tid <  64) { sdata[tid] += sdata[tid +  64]; } __syncthreads(); }

    // Warp shuffle optimization for threads < 32
    if (tid < 32)
    {
        volatile double* smem = sdata;
        if (blockDim.x >=  64) smem[tid] += smem[tid + 32];
        if (blockDim.x >=  32) smem[tid] += smem[tid + 16];
        if (blockDim.x >=  16) smem[tid] += smem[tid +  8];
        if (blockDim.x >=   8) smem[tid] += smem[tid +  4];
        if (blockDim.x >=   4) smem[tid] += smem[tid +  2];
        if (blockDim.x >=   2) smem[tid] += smem[tid +  1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}