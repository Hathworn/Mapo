#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
*
*
* This code is based on the NVIDIA 'reduction' CUDA sample,
* Copyright 1993-2010 NVIDIA Corporation.
*/

extern "C"

__global__ void mismatch(int n, double* actual, double *target, int *mis)
{
    // Use shared memory to reduce global memory access
    __shared__ int local_mis;
    if (threadIdx.x == 0) local_mis = 0;
    __syncthreads();

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        if (target[i] >= 0.5 && actual[i] < 0.5) atomicMax(&local_mis, 1);
        if (target[i] < 0.5 && actual[i] >= 0.5) atomicMax(&local_mis, 1);
    }

    __syncthreads();
    // First thread in the block updates global memory
    if (threadIdx.x == 0 && local_mis == 1) atomicMax(mis, 1);
}