#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize(double *g_idata, double *g_odata, unsigned int n, int maxIndx)
{
    // Load the maximum value from global to shared memory to minimize global memory access
    __shared__ double s_max;
    if (threadIdx.x == 0)
    {
        s_max = g_idata[maxIndx];
    }
    __syncthreads();

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds before accessing global memory
    if (i < n)
    {
        g_odata[i] = exp(g_idata[i] - s_max);
    }
}