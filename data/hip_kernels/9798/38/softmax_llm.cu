#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void softmax(float *g_idata, float *g_odata, unsigned int n)
{
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int gridSize = blockDim.x * 2 * gridDim.x;

    float mySum = 0.0f;

    // Loop unrolling for improved memory coalescing
    while (i < n)
    {
        mySum += expf(g_idata[i]);
        if (i + blockDim.x < n) 
            mySum += expf(g_idata[i + blockDim.x]);
        i += gridSize;
    }

    sdata[tid] = mySum;
    __syncthreads();

    // Optimize reduction using warp shuffles for better warp utilization
    if (blockDim.x >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
    if (blockDim.x >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
    if (blockDim.x >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }

    // Apply warp-level reduction using shuffles
    if (tid < 32)
    {
        volatile float* smem = sdata;
        if (blockDim.x >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; }
        if (blockDim.x >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
        if (blockDim.x >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
        if (blockDim.x >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
        if (blockDim.x >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
        if (blockDim.x >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
    }

    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}