#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel with shared memory access

__global__ void setTargetIndex(int n, double *w, double *out)
{
    // Use shared memory for faster data access
    __shared__ double shared_w[1024];
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data into shared memory
    if (i < n)
    {
        shared_w[threadIdx.x] = w[i];
    }
    __syncthreads();

    // Compute if data is loaded
    if (i < n)
    {
        if (shared_w[threadIdx.x] == 1.0) 
        {
            out[i] -= 1.0;
        }
    }
}