#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized for warp efficiency
extern "C"
__global__ void reduce_exp(double *g_idata, double *g_odata, unsigned int n)
{
    extern __shared__ double sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int gridSize = blockDim.x * 2 * gridDim.x;

    double mySum = 0;

    // Accumulate sums across elements, ensuring memory bounds are respected
    while (i < n)
    {
        mySum += g_idata[i];
        if (i + blockDim.x < n)
            mySum += g_idata[i + blockDim.x];
        i += gridSize;
    }

    // Store the result in shared memory for reduction
    sdata[tid] = mySum;
    __syncthreads();

    // Reduce within shared memory to avoid global memory access
    if (blockDim.x >= 512) { if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads(); }
    if (blockDim.x >= 256) { if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads(); }
    if (blockDim.x >= 128) { if (tid <  64) { sdata[tid] += sdata[tid +  64]; } __syncthreads(); }

    // Warp-synchronous reduction for the last 64 elements
    if (tid < 32)
    {
        volatile double* smem = sdata; // Prevent incorrect compiler optimizations
        if (blockDim.x >=  64) smem[tid] += smem[tid + 32];
        if (blockDim.x >=  32) smem[tid] += smem[tid + 16];
        if (blockDim.x >=  16) smem[tid] += smem[tid +  8];
        if (blockDim.x >=   8) smem[tid] += smem[tid +  4];
        if (blockDim.x >=   4) smem[tid] += smem[tid +  2];
        if (blockDim.x >=   2) smem[tid] += smem[tid +  1];
    }

    // Write block result to global memory
    if (tid == 0)
        g_odata[blockIdx.x] = sdata[0];
}