#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* JCuda - Java bindings for NVIDIA CUDA driver and runtime API
* http://www.jcuda.org
*
*
* This code is based on the NVIDIA 'reduction' CUDA sample,
* Copyright 1993-2010 NVIDIA Corporation.
*/

extern "C"

// Optimize backwardError kernel function
__global__ void backwardError(int n, double *actual, double *target, double* out)
{
    // Calculate index using block and thread indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for faster access
    __shared__ double shared_actual[256]; 
    __shared__ double shared_target[256];

    if (i < n) {
        // Load data into shared memory
        shared_actual[threadIdx.x] = actual[i];
        shared_target[threadIdx.x] = target[i];
        __syncthreads(); // Synchronize threads

        // Perform computation
        out[i] += (shared_actual[threadIdx.x] - shared_target[threadIdx.x]);
    }
}