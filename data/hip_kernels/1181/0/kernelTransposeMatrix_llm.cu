#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NUMBER_THREADS 32

float elapsed_time_ms;
int gpudev = 1;

char *dev_mat_in, *dev_mat_out;

// Optimized kernel for matrix transpose
__global__ void kernelTransposeMatrix(const char *mat_in, char *mat_out, unsigned int rows, unsigned int cols) {
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idy = threadIdx.y + blockIdx.y * blockDim.y;

    // Check boundaries and utilize shared memory for optimized memory access
    if (idx < cols && idy < rows) {
        extern __shared__ char tile[];
        unsigned int index_in = idy * cols + idx;
        unsigned int trans_index = idx * rows + idy;

        tile[threadIdx.y * blockDim.x + threadIdx.x] = mat_in[index_in];

        __syncthreads();

        // Write out the transposed data
        mat_out[trans_index] = tile[threadIdx.y * blockDim.x + threadIdx.x];
    }
}