#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(int *A, int *B, int *C) {
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Simple operation, no need for optimizations like shared memory
    C[i] = __mul24(A[i], 10) + B[i]; // Use __mul24 intrinsic for efficient 24-bit integer multiplication
}