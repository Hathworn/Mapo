#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void childKernel(unsigned int parentThreadIndex, float* data) {
    // Cache parent index in shared memory for reuse 
    __shared__ unsigned int cacheParentIdx;
    if (threadIdx.x == 0) {
        cacheParentIdx = parentThreadIndex;
    }
    __syncthreads();

    // Efficient arithmetic operation
    data[threadIdx.x] = cacheParentIdx + 0.1f * threadIdx.x;
}