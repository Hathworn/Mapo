#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inc (int n, float* a) {
    // Use grid stride loop to improve efficiency and handle larger arrays
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (; i < n; i += stride) {
        a[i] += 1;
    }
}