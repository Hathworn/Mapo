#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int n, float *a, float *b, float *sum) {
    // Unrolling the loop for better performance (assuming n is a multiple of 4)
    int i = blockIdx.x * blockDim.x + threadIdx.x * 4;
    if (i + 3 < n) {
        sum[i] = a[i] + b[i];
        sum[i + 1] = a[i + 1] + b[i + 1];
        sum[i + 2] = a[i + 2] + b[i + 2];
        sum[i + 3] = a[i + 3] + b[i + 3];
    }
    // Handle remaining elements
    for (int j = i; j < n && j < i + 4; j++) {
        sum[j] = a[j] + b[j];
    }
}