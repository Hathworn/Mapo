#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_world()
{
    // Use "blockIdx.x" and "blockDim.x" for a unique global thread ID
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Efficiently print message using unique global thread ID
    printf("Hello World from Global Thread %d !\n", global_id);
}