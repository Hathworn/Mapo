#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_centroids(int d, int k, int* counts, double* centroids) {
    // Calculate global thread ID for effective memory access
    int global_id = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Check if within bounds of matrix range
    if (global_id < d * k) {
        int centroid_idx = global_id / d;
        int coord_idx = global_id % d;

        int count = counts[centroid_idx];
        
        // To avoid introducing divide by zero errors
        // If a centroid has no weight, we'll do no normalization
        // This will keep its coordinates defined.
        if (count < 1) {
            count = 1;
        }
        
        double scale = 1.0 / double(count);
        centroids[coord_idx + d * centroid_idx] *= scale;
    }
}