#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void self_dots(int n, int d, double* data, double* dots) {

    // Determine the ID of the current thread
    int global_id = blockDim.x * blockIdx.x + threadIdx.x;

    if (global_id < n) {
        double accumulator = 0.0;

        // Unroll loop to use registers efficiently
        #pragma unroll
        for (int i = 0; i < d; i++) {
            double value = data[i + global_id * d];
            accumulator += value * value;
        }

        dots[global_id] = accumulator;
    }
}