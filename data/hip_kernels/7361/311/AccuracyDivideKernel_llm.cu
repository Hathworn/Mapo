#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Reduce memory access conflicts by using shared memory
    __shared__ float shared_accuracy;
    
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy;
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        shared_accuracy /= N;
        *accuracy = shared_accuracy;
    }
}