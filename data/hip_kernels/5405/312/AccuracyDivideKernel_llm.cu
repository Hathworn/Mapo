#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Perform division in parallel using multiple threads
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid == 0) {  // Only first thread performs the division
        *accuracy /= N;
    }
    // Optional: Add __syncthreads() if further synchronization is needed
}