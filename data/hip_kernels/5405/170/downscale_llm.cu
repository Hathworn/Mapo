#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    // Calculate global thread index with improved readability
    long ii = threadIdx.x + blockDim.x * blockIdx.x;
    ii += (threadIdx.y + blockDim.y * blockIdx.y) * blockDim.x * gridDim.x;
    
    // Early exit for threads out of range
    if (ii >= no_elements) return;
    
    // Efficient iteration using pre-computed scale limit
    const int scale_limit = scale_factor * scale_factor;
    
    // Precompute the base index for gradOutput_data to avoid redundant calculations
    int base_idx = translate_idx_inv(ii, d1, d2, d3, scale_factor, 0, 0);
    
    // Accumulate results from gradOutput_data to gradInput_data
    for (int s = 0; s < scale_limit; s++) {
        int i = s / scale_factor;
        int j = s % scale_factor;
        int ipidx = base_idx + i * d3 * scale_factor + j;  // Compute ipidx efficiently
        gradInput_data[ii] += gradOutput_data[ipidx];
    }
}