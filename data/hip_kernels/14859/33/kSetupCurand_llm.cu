#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate the unique global thread index for each thread
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialize each thread's state with a unique sequence number
    hiprand_init(seed, tidx, 0, &state[tidx]);
}