#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    const unsigned int totalElements = tgtWidth * tgtHeight;
    
    // Loop efficiently over all target elements using grid-stride loop
    for (uint i = idx; i < totalElements; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Fetch source element and assign to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}