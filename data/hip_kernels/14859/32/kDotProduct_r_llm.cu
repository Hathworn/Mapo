#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;

    // Optimize memory access pattern
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        if (eidx < numCols) {
            shmem[threadIdx.x] += a[eidx] * b[eidx];
        }
    }

    __syncthreads();
    
    // Utilize warp shuffle for reduction
    for (int offset = DP_BLOCKSIZE / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            shmem[threadIdx.x] += shmem[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Final reduction and write result
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}