#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void use_ptr3() {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate unique index
    int stride = blockDim.x * gridDim.x; // Calculate stride
    for (int i = idx; i < 100; i += stride) {
        const_ptr[i] = i; // Write to global memory with coalesced access
    }
}