#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void loadOP(float* input, float* output) {
    // Using thread ID to calculate the index for the Load operation
    int idx = threadIdx.x;
    switch (idx) {
        case 0:
            output[0] = Load<DF>(input);
            break;
        case 1:
            output[1] = Load<CA>(input + 1);
            break;
        case 2:
            output[2] = Load<CG>(input + 2);
            break;
        case 3:
            output[3] = Load<CS>(input + 3);
            break;
        case 4:
            output[4] = Load<CV>(input + 4);
            break;
        case 5:
            output[5] = Load<NC>(input + 5);
            break;
        case 6:
            output[6] = Load<NC_CA>(input + 6);
            break;
        case 7:
            output[7] = Load<NC_CG>(input + 7);
            break;
        case 8:
            output[8] = Load<NC_CS>(input + 8);
            break;
        default:
            // Zero output for unused threads
            output[idx] = 0.0f;
            break;
    }
}