#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void build_sequence_length_padding_offset(const int* sequence_length, const int batch_size, const int max_seq_len, int* valid_word_num, int* tmp_mask_offset)
{
    // Calculate global and local thread indices
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize shared memory for cumulative summation
    __shared__ int cumulative_offset;

    // Handle boundary condition and initialize shared memory
    if (tid == 0) {
        cumulative_offset = 0;
        valid_word_num[0] = 0;
    }
    __syncthreads();

    if (tid < batch_size) {
        // Read and cache sequence length to avoid redundant global memory access
        const int seq_len = sequence_length[tid];
        
        // Update the offset in global memory using thread index
        for (int j = 0; j < seq_len; j++) {
            tmp_mask_offset[tid * max_seq_len + j] = cumulative_offset; 
        }

        // Use atomic addition to safely update shared memory
        atomicAdd(&cumulative_offset, max_seq_len - seq_len);
        atomicAdd(valid_word_num, seq_len);
    }
}