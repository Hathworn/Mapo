#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void OneHotKernel(const float* params, int64_t num_features, int embed_size, int batch_size, const int64_t* indices, float* ret) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global thread index

    if (tid < batch_size * embed_size) { // Check boundary condition
        int batch_id = tid / embed_size; // Determine batch index
        int feature_id = tid % embed_size; // Determine feature index
        ret[tid] = params[(int64_t)indices[batch_id] * embed_size + feature_id]; // Fetch the appropriate value
    }
}