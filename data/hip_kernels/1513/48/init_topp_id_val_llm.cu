#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_topp_id_val(int* topp_id_val_buf, int* topp_offset_buf, const int batch_size, const int vocab_size)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    // Optimize offset buffer initialization by using block synchronization for bid 0
    if(bid == 0) {
        for(int i = tid; i < batch_size + 1; i += blockDim.x) {
            topp_offset_buf[i] = i * vocab_size;
        }
    }

    // Optimize id val buffer initialization for all blocks
    int global_tid = bid * vocab_size + tid;
    while (global_tid < (bid + 1) * vocab_size) {
        topp_id_val_buf[global_tid] = tid;
        tid += blockDim.x;
        global_tid = bid * vocab_size + tid;
    }
}
```
