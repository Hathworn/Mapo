#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void topp_initialization_kernel(bool* finished, int* sequence_length, int* word_ids, int* topp_id_val_buf, int* topp_offset_buf, const int batch_size, const int vocab_size, const int start_id)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    // Calculate global thread index
    int global_tid = tid + bid * blockDim.x;

    // Use shared memory or loop unrolling if beneficial (depends on specifics)
    // Initialize topp_offset_buf, finished, sequence_length, word_ids
    for(int i = global_tid; i < batch_size + 1; i+= blockDim.x * gridDim.x)
    {
        if (i < batch_size + 1) topp_offset_buf[i] = i * vocab_size;
        if (i < batch_size) {
            finished[i] = false;
            sequence_length[i] = 0;
            word_ids[i] = start_id;
        }
    }

    // Initialize topp_id_val_buf
    for (int index = global_tid; index < batch_size * vocab_size; index += blockDim.x * gridDim.x)
    {
        topp_id_val_buf[index] = index % vocab_size;
    }
}