#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GatherKernel(const float* params, int64_t num_features, int embed_size, int batch_size, int query_nnz, const int64_t* indices, float* ret) {
    int tid = threadIdx.x, bid = blockIdx.x;

    extern __shared__ int shmem_indices[];

    // Load indices into shared memory using strides for coalesced access
    for (int i = tid; i < query_nnz; i += blockDim.x) {
        shmem_indices[i] = indices[query_nnz * bid + i];
    }
    __syncthreads();

    // Loop unrolling for performance improvement assuming embed_size is a multiple of warp size
    for (int i = 0; i < query_nnz; ++i) {
        ret[(bid * query_nnz + i) * embed_size + tid] = params[shmem_indices[i] * embed_size + tid];
    }
}