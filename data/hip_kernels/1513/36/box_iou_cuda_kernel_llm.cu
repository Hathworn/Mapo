#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void box_iou_cuda_kernel(float *box_iou, float4 *box1, float4 *box2, long M, long N, int idxJump) {

    // Calculate global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop over stride to alleviate thread divergence and utilize the GPU efficiently
    for (long i = idx; i < M * N; i += idxJump) {

        size_t b1_idx = i / N;
        size_t b2_idx = i % N;

        // Load box coordinates from global memory
        float4 b1 = box1[b1_idx];
        float4 b2 = box2[b2_idx];

        // Compute intersection box coordinates
        float x_tl = fmaxf(b1.x, b2.x);
        float y_tl = fmaxf(b1.y, b2.y);
        float x_br = fminf(b1.z, b2.z);
        float y_br = fminf(b1.w, b2.w);

        // Compute intersection width and height
        float w = fmaxf(0.0f, x_br - x_tl + 1);
        float h = fmaxf(0.0f, y_br - y_tl + 1);

        // Compute intersection area
        float inter = w * h;

        // Compute both box areas
        float area1 = (b1.z - b1.x + 1) * (b1.w - b1.y + 1);
        float area2 = (b2.z - b2.x + 1) * (b2.w - b2.y + 1);

        // Calculate IoU
        float iou = inter / (area1 + area2 - inter);

        // Write the result to global memory
        box_iou[b1_idx * N + b2_idx] = iou;
    }
}