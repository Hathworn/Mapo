#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "box_iou_cuda_kernel_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *box_iou = NULL;
hipMalloc(&box_iou, XSIZE*YSIZE);
float4 *box1 = NULL;
hipMalloc(&box1, XSIZE*YSIZE);
float4 *box2 = NULL;
hipMalloc(&box2, XSIZE*YSIZE);
long M = 2;
long N = XSIZE*YSIZE;
int idxJump = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
box_iou_cuda_kernel<<<gridBlock, threadBlock>>>(box_iou,box1,box2,M,N,idxJump);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
box_iou_cuda_kernel<<<gridBlock, threadBlock>>>(box_iou,box1,box2,M,N,idxJump);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
box_iou_cuda_kernel<<<gridBlock, threadBlock>>>(box_iou,box1,box2,M,N,idxJump);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}