#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "box_encode_kernel.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *targets_dx = NULL;
hipMalloc(&targets_dx, XSIZE*YSIZE);
float *targets_dy = NULL;
hipMalloc(&targets_dy, XSIZE*YSIZE);
float *targets_dw = NULL;
hipMalloc(&targets_dw, XSIZE*YSIZE);
float *targets_dh = NULL;
hipMalloc(&targets_dh, XSIZE*YSIZE);
float4 *boxes = NULL;
hipMalloc(&boxes, XSIZE*YSIZE);
float4 *anchors = NULL;
hipMalloc(&anchors, XSIZE*YSIZE);
float wx = 1;
float wy = 1;
float ww = 1;
float wh = 1;
size_t gt = 1;
size_t idxJump = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
box_encode_kernel<<<gridBlock, threadBlock>>>(targets_dx,targets_dy,targets_dw,targets_dh,boxes,anchors,wx,wy,ww,wh,gt,idxJump);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
box_encode_kernel<<<gridBlock, threadBlock>>>(targets_dx,targets_dy,targets_dw,targets_dh,boxes,anchors,wx,wy,ww,wh,gt,idxJump);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
box_encode_kernel<<<gridBlock, threadBlock>>>(targets_dx,targets_dy,targets_dw,targets_dh,boxes,anchors,wx,wy,ww,wh,gt,idxJump);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}