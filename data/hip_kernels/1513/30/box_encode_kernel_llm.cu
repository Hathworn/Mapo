#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void box_encode_kernel(float *targets_dx, float *targets_dy, float *targets_dw, float *targets_dh, float4 *boxes, float4 *anchors, float wx, float wy, float ww, float wh, size_t gt, size_t idxJump) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for faster repeated access
    __shared__ float4 s_anchors[BLOCK_SIZE];
    __shared__ float4 s_boxes[BLOCK_SIZE];

    while (idx < gt) {
        // Load data into shared memory
        s_anchors[threadIdx.x] = anchors[idx];
        s_boxes[threadIdx.x] = boxes[idx];
        __syncthreads();  // Ensure all threads have loaded data
        
        // Read from shared memory
        float anchors_x1 = s_anchors[threadIdx.x].x;
        float anchors_y1 = s_anchors[threadIdx.x].y;
        float anchors_x2 = s_anchors[threadIdx.x].z;
        float anchors_y2 = s_anchors[threadIdx.x].w;

        float boxes_x1 = s_boxes[threadIdx.x].x;
        float boxes_y1 = s_boxes[threadIdx.x].y;
        float boxes_x2 = s_boxes[threadIdx.x].z;
        float boxes_y2 = s_boxes[threadIdx.x].w;

        float ex_w = anchors_x2 - anchors_x1 + 1;
        float ex_h = anchors_y2 - anchors_y1 + 1;
        float ex_ctr_x = anchors_x1 + 0.5f * ex_w;
        float ex_ctr_y = anchors_y1 + 0.5f * ex_h;

        float gt_w = boxes_x2 - boxes_x1 + 1;
        float gt_h = boxes_y2 - boxes_y1 + 1;
        float gt_ctr_x = boxes_x1 + 0.5f * gt_w;
        float gt_ctr_y = boxes_y1 + 0.5f * gt_h;

        // Reuse previously computed results
        float inv_ex_w = 1.0f / ex_w;
        float inv_ex_h = 1.0f / ex_h;

        targets_dx[idx] = wx * (gt_ctr_x - ex_ctr_x) * inv_ex_w;
        targets_dy[idx] = wy * (gt_ctr_y - ex_ctr_y) * inv_ex_h;
        targets_dw[idx] = ww * logf(gt_w * inv_ex_w);
        targets_dh[idx] = wh * logf(gt_h * inv_ex_h);

        idx += idxJump;
    }
}