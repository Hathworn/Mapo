#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GatherBackwardFuseSgdKernel(const float* grads, int64_t num_features, int embed_size, int batch_size, int query_nnz, const int64_t* indices, float lr, float* params) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int threadID = bid * blockDim.x + tid;  // Unique ID for each thread

    extern __shared__ int64_t shmem_indices[];

    // Coalesced memory access for better performance
    for (int i = tid; i < query_nnz; i += blockDim.x) {
        shmem_indices[i] = indices[query_nnz * bid + i];
    }
    __syncthreads();

    if (tid < embed_size) {  // Ensure only threads needed proceed
        for (int i = 0; i < query_nnz; ++i) {
            // Efficient atomic operation
            atomicAdd(&params[shmem_indices[i] * embed_size + tid],
                      -lr * grads[(bid * query_nnz + i) * embed_size + tid]);
        }
    }
}