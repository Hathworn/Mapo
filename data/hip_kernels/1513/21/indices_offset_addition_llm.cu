#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void indices_offset_addition(int64_t *indices, int64_t *offsets, int64_t *output_indices, int batch_size) {
    const int fea_count = 26;
    __shared__ int64_t smem_offsets[fea_count];

    // Load offsets into shared memory
    if (threadIdx.x < fea_count) {
        smem_offsets[threadIdx.x] = offsets[threadIdx.x];
    }
    __syncthreads();

    // Compute global index and stride value for efficient iteration
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    // Loop through indices with step size equal to total threads
    for (int i = index; i < (batch_size * fea_count); i += stride) {
        output_indices[i] = indices[i] + smem_offsets[i % fea_count];
    }
}