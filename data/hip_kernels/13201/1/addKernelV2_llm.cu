#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NOMINMAX

const unsigned int BLOCK_SIZE = 512;

__global__ void addKernelV2(float *c, const float *a, const float *b)
{
    // Calculate the global thread index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds to avoid memory access errors
    if (i < BLOCK_SIZE) {
        c[i] = a[i] + b[i];
    }
}