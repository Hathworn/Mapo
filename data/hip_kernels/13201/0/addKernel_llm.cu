#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NOMINMAX

const unsigned int BLOCK_SIZE = 512;

__global__ void addKernel(float *c, const float *a, const float *b)
{
    // Calculate global index using block and thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (i < BLOCK_SIZE)
    {
        c[i] = a[i] + b[i];
    }
}