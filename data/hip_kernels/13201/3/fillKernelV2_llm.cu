#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NOMINMAX

const unsigned int BLOCK_SIZE = 512;

__global__ void fillKernelV2(float *a, float* b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    double angle = static_cast<double>(i); // Precompute angle
    double sin_val = sin(angle);           // Precompute sin value
    double cos_val = cos(angle);           // Precompute cos value
    a[i] = sin_val * sin_val;              // Use precomputed sin value
    b[i] = cos_val * cos_val;              // Use precomputed cos value
}