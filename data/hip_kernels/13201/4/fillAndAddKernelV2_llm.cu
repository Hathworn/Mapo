#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NOMINMAX

const unsigned int BLOCK_SIZE = 512;

__global__ void fillAndAddKernelV2(float* c, float *a, float* b)
{
    // Calculate the global index for the thread
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // Precompute sin and cos to reduce redundant calculations
    double sinValue = sin((double)i);
    double cosValue = cos((double)i);
    // Use the precomputed values for a and b
    a[i] = sinValue * sinValue;
    b[i] = cosValue * cosValue;
    // Compute the result directly
    c[i] = a[i] + b[i];
}