#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeVertices_kernel(float4* pos, unsigned int width, unsigned int height, float time)
{
    // Optimize to use correct index calculation for CUDA
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary check to avoid out-of-bounds access
    if (x < width && y < height) {
        // Calculate uv coordinates
        float u = x / (float)width * 2.0f - 1.0f;
        float v = y / (float)height * 2.0f - 1.0f;

        // Calculate simple sine wave pattern
        float freq = 4.0f;
        float w = sin(u * freq + time) * cos(v * freq + time) * 0.5f;

        // Write output vertex
        pos[y * width + x] = make_float4(u, w, v, 1.0f);
    }
}