#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SEED
#define BLOCK_SIZE 32

typedef struct _data {
    char * values;
    char * next_values;
    int width;
    int height;
} data;

__global__ void operate(char * source, char * goal, int sizex, int sizey) {
    // Load relevant elements into shared memory with halo
    __shared__ char local[BLOCK_SIZE + MASK_WIDTH - 1][BLOCK_SIZE + MASK_WIDTH - 1];
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    int prim_x = j - MASK_RADIUS;
    int prim_y = i - MASK_RADIUS;

    // Load data into shared memory using halo
    for(int di = threadIdx.y; di < BLOCK_SIZE + MASK_WIDTH - 1; di += BLOCK_SIZE) {
        for(int dj = threadIdx.x; dj < BLOCK_SIZE + MASK_WIDTH - 1; dj += BLOCK_SIZE) {
            int global_i = prim_y + di;
            int global_j = prim_x + dj;
            if(global_i >= 0 && global_i < sizey && global_j >= 0 && global_j < sizex) {
                local[di][dj] = source[global_i * sizex + global_j];
            } else {
                local[di][dj] = '0';
            }
        }
    }
    __syncthreads();

    // Process each cell
    if(i < sizey && j < sizex) {
        int amount = 0;
        for(int l_i = 0; l_i < MASK_WIDTH; ++l_i) {
            for(int l_j = 0; l_j < MASK_WIDTH; ++l_j) {
                if(local[threadIdx.y + l_i][threadIdx.x + l_j] == '1') {
                    amount++;
                }
            }
        }

        if(source[i * sizex + j] == '1') {
            amount--;
            goal[i * sizex + j] = (amount < 2 || amount > 3) ? '0' : '1';
        } else {
            goal[i * sizex + j] = (amount == 3) ? '1' : '0';
        }
    }
}