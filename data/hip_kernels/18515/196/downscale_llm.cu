#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int w = ii % d3;
    ii /= d3;
    int z = ii % d2;
    ii /= d2;
    int y = ii % d1;
    ii /= d1;
    int x = ii;
    w = w * scale_factor + off_x; // Calculate offset for w
    z = z * scale_factor + off_y; // Calculate offset for z
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w; // Compute linear index with offsets
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int w = ii % d3;
    ii /= d3;
    int z = ii % d2;
    ii /= d2;
    int y = ii % d1;
    ii /= d1;
    int x = ii;
    w = w / scale_factor; // Scale down w
    z = z / scale_factor; // Scale down z
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w; // Compute scaled down linear index
}

__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x; // Calculate linear index
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x + threadIdx.y * blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
    if (ii >= no_elements) return;

    // Unroll the nested loops to improve performance
#pragma unroll
    for (int i = 0; i < scale_factor; i++) {
#pragma unroll
        for (int j = 0; j < scale_factor; j++) {
            int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
            gradInput_data[ii] += gradOutput_data[ipidx]; // Accumulate scaled-down values
        }
    }
}