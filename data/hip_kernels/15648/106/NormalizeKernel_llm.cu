#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (i < h && j < w)
    {
        const int pos = i * s + j;

        float scale = normalization_factor[pos];

        // Inline conditional for inverse scale
        float invScale = (scale != 0.0f) ? (1.0f / scale) : 1.0f;

        image[pos] *= invScale;
    }
}