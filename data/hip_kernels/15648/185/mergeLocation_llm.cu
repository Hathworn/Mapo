#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate global thread index
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory to load data for coalesced access
    __shared__ short2 sharedLoc[256];  // This example assumes max 256 threads per block
    if (ptidx < npoints) {
        sharedLoc[threadIdx.x] = loc_[ptidx];

        __syncthreads();  // Synchronize threads to ensure shared memory is loaded

        x[ptidx] = sharedLoc[threadIdx.x].x * scale;
        y[ptidx] = sharedLoc[threadIdx.x].y * scale;
    }
}