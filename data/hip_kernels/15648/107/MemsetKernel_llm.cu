#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global thread ID
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;

    // Check bounds and update image
    if (j < w && i < h) {
        int pos = i * w + j;
        image[pos] = value;
    }
}