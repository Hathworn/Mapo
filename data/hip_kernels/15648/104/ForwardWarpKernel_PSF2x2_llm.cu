#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst) {
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate pixel corners and deltas
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    // Precompute value and original tx, ty
    float value = src[image_row_offset + j];
    int tx = (int)px;
    int ty = (int)py;

    // Optimized bounds checking and accumulation using a loop
    // Collect the offsets and weights for corners
    int offsets[4][2] = {{0, 0}, {-1, 0}, {-1, -1}, {0, -1}};
    float weights[4] = {dx * dy, (1.0f - dx) * dy, (1.0f - dx) * (1.0f - dy), dx * (1.0f - dy)};

    #pragma unroll
    for (int k = 0; k < 4; ++k) {
        int cur_tx = tx + offsets[k][0];
        int cur_ty = ty + offsets[k][1];
        if (cur_tx >= 0 && cur_tx < w && cur_ty >= 0 && cur_ty < h) {
            float weight = weights[k];
            int index = cur_ty * image_stride + cur_tx;
            _atomicAdd(dst + index, value * weight);
            _atomicAdd(normalization_factor + index, weight);
        }
    }
}