#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Calculate the global thread index
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a conditional operator for a more concise boundary check
    if (pos < len) d_res[pos] = d_src[pos] * scale;
}