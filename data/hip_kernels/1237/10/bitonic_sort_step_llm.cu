#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i ^ j; // Calculate sorting partner

    // Only process valid pairs
    if (ixj > i)
    {
        bool ascending = (i & k) == 0; // Determine sort direction

        // Perform sorting based on direction
        if ((ascending && dev_values[i] > dev_values[ixj]) || (!ascending && dev_values[i] < dev_values[ixj]))
        {
            // Swap elements
            float temp = dev_values[i];
            dev_values[i] = dev_values[ixj];
            dev_values[ixj] = temp;
        }
    }
}