#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialize with unique sequence number for each thread
    hiprand_init(seed, tidx, 0, &state[tidx]);
}