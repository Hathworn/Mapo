#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    // Ensure we're within bounds
    if (tx < numCases) {
        const int label = __float2int_rd(labels[tx]);  // Use fast integer rounding
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);  // Use high performance log for single float

        // Determine the correctness probability
        if (fabsf(labelp - maxp) > 1e-6) {  // Use fabsf for float comparison
            correctProbs[tx] = 0.0f;
        } else {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += __float2int_rd(fabsf(probs[i * numCases + tx] - maxp) < 1e-6);
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}