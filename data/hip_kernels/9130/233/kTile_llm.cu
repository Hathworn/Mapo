#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    // Utilize shared memory for better performance when accessing src
    extern __shared__ float sharedSrc[];
    for (uint i = threadIdx.x; i < srcWidth * srcHeight; i += blockDim.x) {
        sharedSrc[i] = src[i];
    }
    __syncthreads();

    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Use shared memory for the source data access
        tgt[i] = sharedSrc[srcY * srcWidth + srcX];
    }
}