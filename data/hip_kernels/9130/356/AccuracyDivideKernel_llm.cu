#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory to minimize global memory access
    __shared__ float shared_accuracy;
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy;
    }
    __syncthreads();

    // Perform division in parallel
    if (threadIdx.x < N) {
        shared_accuracy /= N;
    }
    __syncthreads();
    
    // Write back the result to global memory from thread 0
    if (threadIdx.x == 0) {
        *accuracy = shared_accuracy;
    }
}