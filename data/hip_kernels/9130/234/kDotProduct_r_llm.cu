#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;

    // Initialize shared memory with zero
    shmem[threadIdx.x] = 0;

    // Efficient boundary check and dot product calculation
    if (eidx < numElements) {
        for (uint i = eidx; i < numElements; i += gridDim.x * DP_BLOCKSIZE) {
            shmem[threadIdx.x] += a[i] * b[i];
        }
    }
    __syncthreads();

    // Unrolling tree-based reduction logic
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();
    if (threadIdx.x < 32) {
        // Use volatile to prevent optimization issues in warp reduction
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
        if (threadIdx.x == 0) {
            target[blockIdx.x] = mysh[0];
        }
    }
}