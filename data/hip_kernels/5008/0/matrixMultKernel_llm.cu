#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 1200
#define THREADS 1024

__global__ void matrixMultKernel (double *a, double *b, double *c, int n)
{
    // Thread and block index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    double v = 0;

    // Shared memory for tiles of A and B
    __shared__ double a_tile[32][32];
    __shared__ double b_tile[32][32];

    for (int i = 0; i < n; i += blockDim.x) {
        // Load elements into shared memory
        if (row < n && (i + threadIdx.x) < n)
            a_tile[threadIdx.y][threadIdx.x] = a[row * n + i + threadIdx.x];
        else
            a_tile[threadIdx.y][threadIdx.x] = 0.0;

        if (col < n && (i + threadIdx.y) < n)
            b_tile[threadIdx.y][threadIdx.x] = b[(i + threadIdx.y) * n + col];
        else
            b_tile[threadIdx.y][threadIdx.x] = 0.0;

        // Synchronize to make sure the sub-matrices are loaded
        __syncthreads();

        // Each thread computes one element in the block sub-matrix
        for (int j = 0; j < blockDim.x; j++) {
            v += a_tile[threadIdx.y][j] * b_tile[j][threadIdx.x];
        }

        // Synchronize to ensure computation is done before loading new sub-matrices
        __syncthreads();
    }

    // Write the result
    if (row < n && col < n) {
        c[row * n + col] = v;
    }
}