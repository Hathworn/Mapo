#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(int *a, int *b, int *c, int n) // Pass array size as parameter
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < n) // Use parameter passed for bounds checking
    {
        c[index] = a[index] + b[index];
    }
}
```
