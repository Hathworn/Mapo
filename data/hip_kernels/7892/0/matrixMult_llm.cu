#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/* CUDA API header files */


extern "C"
__global__ void matrixMult(const double *Md, const double *Nd, double *Pd, int size)
{
    // Use shared memory to optimize memory access
    __shared__ double Mds[16][16];
    __shared__ double Nds[16][16];

    int row = blockDim.x * blockIdx.x + threadIdx.x;
    int col = blockDim.y * blockIdx.y + threadIdx.y;
    
    double Pvalue = 0.0;

    for (int k = 0; k < (size + 15) / 16; ++k) {
        // Load Md and Nd into shared memory
        if (k * 16 + threadIdx.x < size && row < size)
            Mds[threadIdx.y][threadIdx.x] = Md[row + (k * 16 + threadIdx.x) * size];
        else
            Mds[threadIdx.y][threadIdx.x] = 0.0;

        if (k * 16 + threadIdx.y < size && col < size)
            Nds[threadIdx.y][threadIdx.x] = Nd[col * size + k * 16 + threadIdx.y];
        else
            Nds[threadIdx.y][threadIdx.x] = 0.0;

        // Synchronize to make sure the sub-matrices are loaded
        __syncthreads();

        // Multiply the loaded matrices
        for (int n = 0; n < 16; ++n)
            Pvalue += Mds[threadIdx.y][n] * Nds[n][threadIdx.x];

        // Synchronize to make sure that computation is done before the next sub-matrix is loaded
        __syncthreads();
    }

    // Write the result
    if (row < size && col < size)
        Pd[col * size + row] = Pvalue;
}