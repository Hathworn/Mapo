#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel() {
    // Use thread and block IDs for potential parallel work distribution
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) { // Ensure only one thread prints
        printf("Hello, world!\n");
    }
}