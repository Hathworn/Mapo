#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addOne(int* array, int size) {
    // Calculate the global index for the current thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure the index is within bounds
    if (idx < size) {
        array[idx]++;
    }
}