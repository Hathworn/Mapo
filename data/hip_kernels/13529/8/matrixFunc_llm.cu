#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixFunc(float *F, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure indices are within bounds
    if (i < size && j < size - 1) {
        // Loop unrolling and leveraging thread parallelism
        #pragma unroll
        for(int k = 0; k < 100; k++) {
            F[i * size + j] = F[(i-1) * size + j + 1] + F[i * size + j + 1];
        }
    }
}