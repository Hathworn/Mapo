#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16 // Add macro for TILE_WIDTH if not defined elsewhere

__global__ void matrixMultiply3(float* A, float* C, int size) {

    float CValue = 0;

    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    __shared__ float As[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bs[TILE_WIDTH][TILE_WIDTH]; // Separate shared memory for B

    for (int k = 0; k < (TILE_WIDTH + size - 1) / TILE_WIDTH; k++) {

        // Load As from global to shared memory
        if (k * TILE_WIDTH + threadIdx.x < size && Row < size)
            As[threadIdx.y][threadIdx.x] = A[Row * size + k * TILE_WIDTH + threadIdx.x];
        else
            As[threadIdx.y][threadIdx.x] = 0.0;

        // Load Bs from global to shared memory
        if (k * TILE_WIDTH + threadIdx.y < size && Col < size) 
            Bs[threadIdx.y][threadIdx.x] = A[(k*TILE_WIDTH + threadIdx.y) * size + Col];
        else
            Bs[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads(); // Ensure all threads have loaded data to shared memory

        // Collaborative computation
        for (int n = 0; n < TILE_WIDTH; ++n)
            CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

        __syncthreads(); // Ensure computation is done before next iteration
    }

    // Write result to global memory
    if (Row < size && Col < size)
        C[Row * size + Col] = CValue;
}