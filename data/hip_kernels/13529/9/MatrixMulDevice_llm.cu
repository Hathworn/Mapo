#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function for matrix multiplication
__global__ void MatrixMulDevice(float *A, float *B, float *C, int *matrixSize)
{
    int chunk = (*matrixSize) / gridDim.x;
    int i, k;
    int sum = 0;

    // Calculate global row index
    int rowStart = blockIdx.x * chunk;
    int rowEnd = rowStart + chunk;

    // Loop through each row assigned to the block
    for(i = rowStart; i < rowEnd; i++) {
        sum = 0;
        
        // Accumulate dot product for a single row
        for(k = 0; k < *matrixSize; k++) {
            sum += A[i * *matrixSize + k] * B[k * *matrixSize + threadIdx.x];
        }

        // Store the result in C
        C[i * *matrixSize + threadIdx.x] = sum;
    }
}