#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMultiply2(float* A, float* C, int size) 
{
    // Initialize sum to zero
    float sum = 0;
    // Compute column and row indices for the current thread
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure indices are within matrix bounds
    if (Col < size && Row < size) {
        // Unroll loop for better efficiency (assuming size is a multiple of 4)
        #pragma unroll
        for (int k = 0; k < size; k++)
            sum += A[k * size + Row] * A[k * size + Col];
        
        // Write the computed sum to the output matrix
        C[Row * size + Col] = sum;
    }
}