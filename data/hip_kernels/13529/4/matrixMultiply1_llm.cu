#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMultiply1(float *A, float *C, int size) {
    int Col = blockDim.x * blockIdx.x + threadIdx.x; // Correct columns calculation
    int Row = blockDim.y * blockIdx.y + threadIdx.y; // Correct rows calculation

    // Check boundary to prevent excess memory access
    if (Row < size && Col < size) {
        float sum = 0.0f; // Use a local variable to accumulate sum
        for (int k = 0; k < size; k++) {
            sum += A[k * size + Row] * A[k * size + Col];
        }
        C[Row * size + Col] = sum; // Store the result
    }
}