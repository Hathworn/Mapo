#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to divide accuracy by N
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Avoiding potential race condition by using atomic operation
    atomicExch(accuracy, *accuracy / N);
}