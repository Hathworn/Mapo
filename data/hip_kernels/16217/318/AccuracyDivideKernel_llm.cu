#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Using a shared memory variable to reduce repetitive global memory access
    __shared__ float local_accuracy;
    if (threadIdx.x == 0) {
        local_accuracy = *accuracy;
        local_accuracy /= N;
        *accuracy = local_accuracy;
    }
}