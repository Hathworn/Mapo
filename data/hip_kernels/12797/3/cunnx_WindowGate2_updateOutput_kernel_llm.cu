#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_WindowGate2_updateOutput_kernel( float *output, float *centroids, float *normalizedCentroids, float *inputIndice, float *outputIndice, const float *input, const float *noise, int inputSize, int outputSize, int inputWindowSize, int outputWindowSize, int windowStride, int train)
{
    __shared__ float buffer[WINDOWGATE2_THREADS + 1];
    unsigned int tx = threadIdx.x;
    unsigned int k = blockIdx.x;
    const float *input_k = input + inputSize * k;
    float *output_k = output + outputWindowSize * k;

    // Initialize buffer
    float sum = 0;
    for (unsigned int i = tx; i < inputSize; i += blockDim.x)
        sum += input_k[i] * (float)(i + 1);
    
    buffer[tx] = sum;

    // Optimized reduction
    __syncthreads();
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tx < stride)
            buffer[tx] += buffer[tx + stride];
        __syncthreads();
    }

    if (tx == 0)
    {
        float centroid = buffer[0] / (float)inputSize;
        normalizedCentroids[k] = centroid;

        if (train)
        {
            centroid += noise[k];
            centroid = fminf(fmaxf(0, centroid), 1);
        }

        centroid *= (float)outputSize;

        float inputIdx = centroid / (float)inputSize - 0.5f * (float)inputWindowSize;
        float outputIdx = centroid - 0.5f * (float)outputWindowSize;

        inputIdx = fminf(fmaxf(1, inputIdx), inputSize - inputWindowSize + 1);
        outputIdx = fminf(fmaxf(1, outputIdx), outputSize - outputWindowSize + 1);

        inputIdx = ceilf(inputIdx);
        outputIdx = ceilf(outputIdx);

        centroid -= (outputIdx - 1);

        inputIndice[k] = (int)inputIdx;
        outputIndice[k] = (int)outputIdx;
        centroids[k] = centroid;

        buffer[WINDOWGATE2_THREADS] = inputIdx;
    }

    __syncthreads();

    float inputIdx = buffer[WINDOWGATE2_THREADS];
    const float *inputWindow = input_k + (int)inputIdx;

    for (int i = tx; i < outputWindowSize; i += blockDim.x)
    {
        output_k[i] = inputWindow[(int)floorf(((float)i) / windowStride)];
    }
}