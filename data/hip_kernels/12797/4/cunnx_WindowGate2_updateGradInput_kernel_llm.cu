#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_WindowGate2_updateGradInput_kernel(
    float *gradInput, float *error, float* targetCentroids, 
    const float *centroids, const float *input, const float *inputIndice, 
    const float *outputIndice, const float* output, const float* gradOutput, 
    int inputSize, int outputSize, int inputWindowSize, int outputWindowSize, 
    int windowStride, float c, float d, float e, float lr)
{
    unsigned int tx = threadIdx.x;
    unsigned int k = blockIdx.x;
    const float *gradOutput_k = gradOutput + outputWindowSize * k;
    float *gradInput_k = gradInput + inputSize * k;

    float *gradInputWindow = gradInput_k + static_cast<int>(inputIndice[k] - 1);

    // Use shared memory for partial sums
    __shared__ float sharedSum[256];

    float sum = 0;
    for (int i = tx; i < inputWindowSize; i += blockDim.x) {
        const float *gradOutputChannel = gradOutput_k + i * windowStride;
        for (int j = 0; j < windowStride; j++) {
            sum += gradOutputChannel[j];
        }
        sharedSum[tx] = sum;  // Store the result in shared memory

        __syncthreads();  // Ensure all threads have written their sums

        // Sum within the shared memory array
        if (tx == 0) {
            float blockSum = 0;
            for (int j = 0; j < blockDim.x; j++) {
                blockSum += sharedSum[j];
            }
            gradInputWindow[i] += blockSum;
        }
        
        __syncthreads();  // Ensure the first thread updates the memory
    }
}