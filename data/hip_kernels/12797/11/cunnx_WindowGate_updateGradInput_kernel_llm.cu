#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_WindowGate_updateGradInput_kernel(float *gradInput, float *error, float* targetCentroids, const float *centroids, const float *input, const float *outputIndice, const float* output, const float* gradOutput, int inputSize, int outputSize, int outputWindowSize, float c, float d, float e, float lr) {
    __shared__ float buffer[WINDOWGATE_THREADS + 1];
    unsigned int tx = threadIdx.x;
    unsigned int k = blockIdx.x;
    const float *gradOutput_k = gradOutput + outputWindowSize * k;
    const float *output_k = output + outputWindowSize * k;
    const float *input_k = input + inputSize * k;
    float *gradInput_k = gradInput + inputSize * k;
    float centroid = centroids[k];

    // Initialize shared buffer for accumulation
    buffer[tx] = 0.0f;

    // Compute gradient of centroid in parallel
    for (unsigned int i = tx; i < outputWindowSize; i += blockDim.x) {
        buffer[tx] += gradOutput_k[i] * output_k[i] * ((float)(i + 1) - centroid);
    }

    // Perform parallel reduction
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (tx < stride) {
            buffer[tx] += buffer[tx + stride];
        }
    }

    if (tx == 0) {
        int outputIdx = outputIndice[k];
        float gradCentroid = buffer[0] * c;
        centroid -= (lr * gradCentroid);
        centroid += outputIdx - 1;
        centroid /= (float)(outputSize);
        targetCentroids[k] = centroid;
        buffer[WINDOWGATE_THREADS] = centroid * (float)(inputSize);
    }

    __syncthreads();
    float targetCentroid = buffer[WINDOWGATE_THREADS];

    // Reinitialize buffer for second operation
    buffer[tx] = 0.0f;

    // Calculate target and gradients for each input element
    for (int i = tx; i < inputSize; i += blockDim.x) {
        float target = (float)(i + 1) - targetCentroid;
        target = d * expf(target * target * e);
        float inputVal = input_k[i];
        buffer[tx] -= logf(inputVal + 1e-7f) * target;  // Prevent log(0)
        gradInput_k[i] = -target / (inputVal + 1e-7f);
    }

    // Perform parallel reduction for error accumulation
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (tx < stride) {
            buffer[tx] += buffer[tx + stride];
        }
    }

    if (tx == 0) {
        error[k] = buffer[0];
    }
}