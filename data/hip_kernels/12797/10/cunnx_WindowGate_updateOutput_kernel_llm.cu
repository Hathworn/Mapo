#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_WindowGate_updateOutput_kernel( float *output, float *centroids, float *normalizedCentroids, float *outputIndice, const float *input, const float *noise, int inputSize, int outputSize, int outputWindowSize, float a, float b, int train)
{
    __shared__ float buffer[WINDOWGATE_THREADS];
    unsigned int tx = threadIdx.x;
    unsigned int k = blockIdx.x;
    const float *input_k = input + inputSize*k;
    float *output_k = output + outputWindowSize*k;

    // Initialize buffer to zero
    buffer[tx] = 0.0f;
    
    // Sum up the input values weighted by their indices
    for (unsigned int i = tx; i < inputSize; i += blockDim.x) {
        buffer[tx] += input_k[i] * (float)(i + 1);
    }

    // Block reduction to compute sum
    for (unsigned int stride = WINDOWGATE_THREADS >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (tx < stride) {
            buffer[tx] += buffer[tx + stride];
        }
    }

    // Compute centroid and output index if thread index is 0
    if (tx == 0) {
        float centroid = buffer[0] / (float)inputSize;  // Normalize centroid
        normalizedCentroids[k] = centroid;

        if (train) {
            centroid += noise[k];
            centroid = fminf(fmaxf(0.0f, centroid), 1.0f);  // Clamp centroid
        }

        centroid *= (float)outputSize;  // Align centroid to output size

        // Calculate output index and ensure it's within valid bounds
        float outputIdx = centroid - 0.5f * (float)outputWindowSize;
        outputIdx = fminf(outputIdx, (float)(outputSize - outputWindowSize + 1));
        outputIdx = fmaxf(outputIdx, 1.0f);
        outputIdx = ceilf(outputIdx);  // Round up to nearest integer
        
        centroid -= (outputIdx - 1.0f);  // Align centroid to outputWindow
        outputIndice[k] = (int)outputIdx;
        centroids[k] = centroid;
        buffer[0] = centroid;
    }

    __syncthreads();
    
    float centroid = buffer[0];

    // Apply Gaussian blur
    for (int i = tx; i < outputWindowSize; i += blockDim.x) {
        float x = (float)(i + 1) - centroid;
        output_k[i] = a * expf(x * x * b);  // Compute output value
    }
}