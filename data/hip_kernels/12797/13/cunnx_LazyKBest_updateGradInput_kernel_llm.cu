#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_LazyKBest_updateGradInput_kernel(float *gradInput, const float *indice, const float *gradOutput, int inputSize, int outputSize)
{
    int tx = threadIdx.x + blockIdx.x * blockDim.x;  // Use unique global thread index
    int step = blockDim.x * gridDim.x;               // Adjust step for all blocks

    float *gradInput_k = gradInput + blockIdx.x * inputSize;
    const float *gradOutput_k = gradOutput + blockIdx.x * outputSize;
    const float *indice_k = indice + blockIdx.x * outputSize;

    for (int i = tx; i < outputSize; i += step)
    {
        int index = static_cast<int>(indice_k[i] - 1); // Cache index for reuse
        gradInput_k[index] = gradOutput_k[i];
    }
}