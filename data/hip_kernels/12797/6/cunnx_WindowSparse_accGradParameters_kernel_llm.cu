#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_WindowSparse_accGradParameters_kernel( float *gradWeight, float* gradBias, float *gradOutput, float *input, float *inputIndice, float *outputIndice, int inputWindowSize, int outputWindowSize, int inputSize, int outputSize, float scale)
{
    __shared__ float buffer[WINDOWSPARSE_THREADS];
    int tx = threadIdx.x;
    int i_step = blockDim.x;
    int k = blockIdx.x;

    int inputIdx = (int)inputIndice[k] - 1;
    int outputIdx = (int)outputIndice[k] - 1;

    float *input_k = input + k * inputWindowSize;
    float *gradOutput_k = gradOutput + k * outputWindowSize;
    float *gradWeight_k = gradWeight + outputIdx * inputSize + inputIdx;
    float *gradBias_k = gradBias + outputIdx;

    // Preload input to shared memory to reduce global memory accesses
    if (tx < inputWindowSize) {
        buffer[tx] = input_k[tx] * scale;
    }
    __syncthreads();

    // Iterate over input window size and update weights
    for (int i = tx; i < inputWindowSize; i += i_step) {
        float buffer_val = buffer[i]; // Load once from shared memory
        for (int j = 0; j < outputWindowSize; ++j) {
            atomicAdd(&(gradWeight_k[j * inputSize + i]), gradOutput_k[j] * buffer_val);
        }
    }

    // Calculate and update biases
    for (int j = tx; j < outputWindowSize; j += i_step) {
        atomicAdd(&(gradBias_k[j]), gradOutput_k[j] * scale);
    }
}
```
