#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_BlockSparse_updateOutput_kernel(float *output, const float *input, const float *outputIndice, const float *outputScale, const float *bias, int outputSize, int nOutputBlock, int inputWindowSize, int outputWindowSize)
{
    __shared__ float buffer[BLOCKSPARSE_THREADS];
    int tx = threadIdx.x;
    int k = blockIdx.x;

    float *output_k = output + k * outputWindowSize * outputSize;
    const float *input_k = input + k * inputWindowSize * outputWindowSize * outputSize;
    const float *outputIndice_k = outputIndice + k * outputWindowSize;
    const float *outputScale_k = outputScale + k * outputWindowSize;

    for (int m = 0; m < outputWindowSize; m++)
    {
        int outputIdx = (int)outputIndice_k[m] - 1;
        float outputScale = outputScale_k[m];

        // Loop unrolling and reducing number of operations
        int j = tx;
        while (j < outputSize)
        {
            float val = bias[outputIdx * outputSize + j];

            for (int l = 0; l < inputWindowSize; l++)
            {
                val += input_k[l * outputWindowSize * outputSize + m * outputSize + j];
            }

            output_k[m * outputSize + j] = outputScale * val;
            j += blockDim.x; // Increment by block size to reduce control divergence
        }
    }
}