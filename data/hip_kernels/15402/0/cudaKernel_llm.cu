#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Copyright (c) 2019, NVIDIA Corporation. All rights reserved.
// This work is made available under the Nvidia Source Code License-NC.
// To view a copy of this license, visit
// https://nvlabs.github.io/stylegan2/license.html

__global__ void cudaKernel(void)
{
    // Use printf strings with __syncthreads for potential printf overlap mitigation
    printf("GPU says hello from block %d, thread %d.\n", blockIdx.x, threadIdx.x);
}