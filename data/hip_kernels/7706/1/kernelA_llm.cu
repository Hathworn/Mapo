#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelA(float* r, float* x, float* y, float* z, int size)
{
    // Calculate global index based on block and thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index is within bounds
    if (i < size) {
        r[i] = x[i] * y[i] + z[i];
    }
}