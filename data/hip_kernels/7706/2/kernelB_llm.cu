#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
    // Calculate global thread index
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize loop with a single calculation using the global index
    if (globalIdx < size) {
        r[globalIdx] = x[globalIdx] * y[globalIdx] + z[globalIdx];
    }
}