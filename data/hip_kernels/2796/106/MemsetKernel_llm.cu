#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Compute global thread index in x and y directions
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Bounds check to ensure index is within limits
    if (i < h && j < w) {
        // Compute linear index and set value
        const int pos = i * w + j;
        image[pos] = value;
    }
}