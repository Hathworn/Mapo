#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate subpixel positions and weights
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    int value_idx = image_row_offset + j;
    float value = src[value_idx];

    // Precompute target pixel positions and weights
    int coords[4][2] = {{(int)px, (int)py}, {(int)px - 1, (int)py}, {(int)px - 1, (int)py - 1}, {(int)px, (int)py - 1}};
    float weights[4] = {dx * dy, (1.0f - dx) * dy, (1.0f - dx) * (1.0f - dy), dx * (1.0f - dy)};

    // Iterate over all corners and update accumulation
    for (int n = 0; n < 4; ++n) {
        int tx = coords[n][0];
        int ty = coords[n][1];
        if (tx >= 0 && tx < w && ty >= 0 && ty < h) { // Simplified bounds check
            float weight = weights[n];
            int target_idx = ty * image_stride + tx;
            _atomicAdd(dst + target_idx, value * weight);
            _atomicAdd(normalization_factor + target_idx, weight);
        }
    }
}