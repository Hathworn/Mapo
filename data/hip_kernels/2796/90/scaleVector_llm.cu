#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Calculate position based on block and thread indices
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if position is within bounds and perform scaling
    if (pos < len) {
        d_res[pos] = d_src[pos] * scale;
    }
}