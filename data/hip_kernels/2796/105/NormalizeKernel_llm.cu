#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image) {
    // Calculate the global index for the current thread
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds to ensure the thread is within the image dimensions
    if (i < h && j < w) {
        const int pos = i * s + j; // Compute the linear index for the image element

        float scale = normalization_factor[pos]; // Load normalization factor

        // Compute the inverse scale, checking for zero to avoid division by zero
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

        // Normalize the image pixel value by the inverse scale
        image[pos] *= invScale;
    }
}