#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    // Calculate global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for better performance
    __shared__ float shared_a[256];
    
    // Load data into shared memory if within bounds
    if (idx < N)
    {
        shared_a[threadIdx.x] = a[idx];

        // Synchronize threads to ensure all data is loaded before proceeding
        __syncthreads();

        // Perform computation using shared memory
        shared_a[threadIdx.x] *= shared_a[threadIdx.x];

        // Synchronize threads before writing back to global memory
        __syncthreads();

        // Write results back to global memory
        a[idx] = shared_a[threadIdx.x];
    }
}