#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_rand_init(hiprandState *__restrict__ pState, int seed) {
const int gtid_x = blockIdx.x * blockDim.x + threadIdx.x;
const int gtid_y = blockIdx.y * blockDim.y + threadIdx.y;
const int gtid = gtid_y * gridDim.x * blockDim.x + gtid_x;
hiprandState state;
hiprand_init(seed, gtid, 0, &state);
pState[gtid] = state;
}