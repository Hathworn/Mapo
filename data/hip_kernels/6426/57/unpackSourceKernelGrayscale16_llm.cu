#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unpackSourceKernelGrayscale16(uint16_t* __restrict__ dst, unsigned pitch, const hipSurfaceObject_t src, unsigned width, unsigned height) {
    // Pre-calculate indices
    const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds only once
    if (x < width && y < height) {
        float val;
        // Coalesced access and use shared memory for surf2Dread if possible
        __shared__ float shared_val[BLOCK_SIZE_X][BLOCK_SIZE_Y]; // Assume BLOCK_SIZE_X and BLOCK_SIZE_Y are defined
        surf2Dread(&shared_val[threadIdx.y][threadIdx.x], src, x * sizeof(float), y);
        __syncthreads(); // Ensure all threads read the surface before proceeding

        // Convert directly without intermediate steps
        const uint16_t u16 = __float2half_rn(fmaxf(0.f, fminf((float)USHRT_MAX, shared_val[threadIdx.y][threadIdx.x] * 1000.f)));
        // Reduce global memory write by using a register
        dst[y * pitch + x] = u16;
    }
}