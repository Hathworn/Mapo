#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce6(const float* g_idata, float* g_odata, float* g_omask, unsigned int n) {
    extern __shared__ float sharedData[];
    float* sdata = &sharedData[0];
    float* smask = &sharedData[blockDim.x];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int gridSize = blockDim.x * 2 * gridDim.x;
    sdata[tid] = 0.0f; // Initialize shared memory
    smask[tid] = 0.0f;

    // Optimize for memory coalescing
    while (i < n) {
        sdata[tid] += g_idata[i];
        smask[tid] += (g_idata[i] > 0 ? 1.0f : 0.0f);
        if (i + blockDim.x < n) {
            sdata[tid] += g_idata[i + blockDim.x];
            smask[tid] += (g_idata[i + blockDim.x] > 0 ? 1.0f : 0.0f);
        }
        i += gridSize;
    }
    __syncthreads();

    // Unroll the loop for warp efficiency
    if (blockDim.x >= 512) {
        if (tid < 256) {
            sdata[tid] += sdata[tid + 256];
            smask[tid] += smask[tid + 256];
        }
        __syncthreads();
    }
    if (blockDim.x >= 256) {
        if (tid < 128) {
            sdata[tid] += sdata[tid + 128];
            smask[tid] += smask[tid + 128];
        }
        __syncthreads();
    }
    if (blockDim.x >= 128) {
        if (tid < 64) {
            sdata[tid] += sdata[tid + 64];
            smask[tid] += smask[tid + 64];
        }
        __syncthreads();
    }

    // Use warp-level operations without __syncthreads
    if (tid < 32) {
        if (blockDim.x >= 64) {
            sdata[tid] += sdata[tid + 32];
            smask[tid] += smask[tid + 32];
        }
        if (blockDim.x >= 32) {
            sdata[tid] += sdata[tid + 16];
            smask[tid] += smask[tid + 16];
        }
        if (blockDim.x >= 16) {
            sdata[tid] += sdata[tid + 8];
            smask[tid] += smask[tid + 8];
        }
        if (blockDim.x >= 8) {
            sdata[tid] += sdata[tid + 4];
            smask[tid] += smask[tid + 4];
        }
        if (blockDim.x >= 4) {
            sdata[tid] += sdata[tid + 2];
            smask[tid] += smask[tid + 2];
        }
        if (blockDim.x >= 2) {
            sdata[tid] += sdata[tid + 1];
            smask[tid] += smask[tid + 1];
        }
    }

    // Write block result to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
        g_omask[blockIdx.x] = smask[0];
    }
}