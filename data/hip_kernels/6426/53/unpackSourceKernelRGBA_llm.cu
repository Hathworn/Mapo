#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unpackSourceKernelRGBA(uint32_t* dst, unsigned pitch, const hipSurfaceObject_t src, unsigned width, unsigned height) {
    // Compute x and y indices for the thread
    const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure the indices are within the image dimensions
    if (x < width && y < height) {
        uint32_t val;

        // Use efficient surface read operation for fetching data
        surf2Dread(&val, src, x * sizeof(uint32_t), y);

        // Store the fetched value in the destination array
        dst[y * pitch + x] = val;
    }
}