#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void onBothBufferOperatorKernel(const int warpWidth, const int input0OffsetX, const int input0OffsetY, const int input0Width, const int input0Height, const uint32_t* input0Buffer, const int input1OffsetX, const int input1OffsetY, const int input1Width, const int input1Height, const uint32_t* input1Buffer, const int outputOffsetX, const int outputOffsetY, const int outputWidth, const int outputHeight, uint32_t* outputMask) {
    // Thread indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure within bounds of the output
    if (x >= outputWidth || y >= outputHeight) return;

    uint32_t v = 0;
    const int outputX = x + outputOffsetX;
    const int outputY = y + outputOffsetY;
    // Calculating input indices with modulo optimization for wrapping
    const int input0X = (outputX + warpWidth - input0OffsetX) % warpWidth;
    const int input0Y = outputY - input0OffsetY;
    const int input1X = (outputX + warpWidth - input1OffsetX) % warpWidth;
    const int input1Y = outputY - input1OffsetY;

    // Check bounds and perform operations efficiently
    if (input1X >= 0 && input1X < input1Width &&
        input1Y >= 0 && input1Y < input1Height &&
        input0X >= 0 && input0X < input0Width &&
        input0Y >= 0 && input0Y < input0Height) {
        v = (input0Buffer[input0Y * input0Width + input0X] > 0 && 
             input1Buffer[input1Y * input1Width + input1X] > 0) ? 1 : 0;
    }

    // Assign the computed value to the output mask
    outputMask[y * outputWidth + x] = v;
}