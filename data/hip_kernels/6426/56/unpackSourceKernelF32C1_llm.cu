#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void unpackSourceKernelF32C1(float* dst, unsigned pitch, const hipSurfaceObject_t src, unsigned width, unsigned height) {
    const unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        // Optimize memory access by coalescing writes
        float val;
        surf2Dread(&val, src, x * sizeof(float), y);
        dst[y * pitch + x] = val;
    }
}