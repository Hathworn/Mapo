#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "unpackSourceKernelF32C1_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *dst = NULL;
hipMalloc(&dst, XSIZE*YSIZE);
unsigned pitch = 2;
const hipSurfaceObject_t src = 1;
unsigned width = XSIZE;
unsigned height = YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
unpackSourceKernelF32C1<<<gridBlock, threadBlock>>>(dst,pitch,src,width,height);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
unpackSourceKernelF32C1<<<gridBlock, threadBlock>>>(dst,pitch,src,width,height);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
unpackSourceKernelF32C1<<<gridBlock, threadBlock>>>(dst,pitch,src,width,height);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}