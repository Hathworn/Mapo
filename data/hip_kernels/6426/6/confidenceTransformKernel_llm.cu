#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void confidenceTransformKernel(const int width, const int height, const float threshold, const float gamma, const float clampedValue, const float* inputConfidence, float* outputConfidence) {
    // Calculate global thread coordinates
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Exit if thread is out of bounds
    if (x >= width || y >= height) return;
    
    // Compute flattened index
    uint32_t index = y * width + x;
    
    // Load input value from global memory
    float inputValue = inputConfidence[index];
    
    // Apply confidence transformation
    outputConfidence[index] = (inputValue < threshold) ? 0.0f : powf(inputValue, gamma);
}