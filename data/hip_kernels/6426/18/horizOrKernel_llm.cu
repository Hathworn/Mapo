#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void horizOrKernel(const uint32_t* __restrict__ contrib, uint32_t* __restrict__ rowHasImage, unsigned panoWidth, unsigned panoHeight) {
    unsigned row = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load input only if row is within panoHeight
    if (row < panoHeight) {
        const uint32_t* rowp = contrib + panoWidth * row;
        uint32_t accum = 0;

        // Unroll loop for better performance
        for (unsigned col = 0; col < panoWidth; col += 4) {
            accum |= rowp[col];
            if (col + 1 < panoWidth) accum |= rowp[col + 1];
            if (col + 2 < panoWidth) accum |= rowp[col + 2];
            if (col + 3 < panoWidth) accum |= rowp[col + 3];
        }
        
        // Store the accumulated result
        rowHasImage[row] = accum;
    }
}