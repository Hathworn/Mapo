#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void resetArrayKernel(hipSurfaceObject_t dst, size_t width, size_t height) {
    // Calculate global thread ID for x and y dimensions
    unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread accesses are within bounds
    if (x < width && y < height) {
        // Optimize memory write with warp-level memory operation (if applicable)
        surf2Dwrite(0, dst, x * sizeof(uint32_t), y);
    }
}