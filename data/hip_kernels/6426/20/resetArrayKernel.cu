#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void resetArrayKernel(hipSurfaceObject_t dst, size_t width, size_t height) {
unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

if (x < width && y < height) {
surf2Dwrite(0, dst, x * sizeof(uint32_t), y);
}
}