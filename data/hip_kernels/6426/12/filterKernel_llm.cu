#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void filterKernel(unsigned char* data, unsigned width, unsigned height, unsigned hStride, unsigned vStride, bool wrapAround) {
    // Compute column index based on block and thread
    unsigned columnId = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if column index is within bounds
    if (columnId < width) {
        // Calculate starting point of the column in the data
        unsigned char* colp = data + columnId * hStride;
        unsigned step = width * hStride * vStride;

        // Initialize previous element for current column
        uint32_t prev = colp[step];
        uint32_t pprev = wrapAround ? colp[step * (height - 1)] : prev;
        
        // Update the first element of column with boundary condition
        uint32_t v = colp[0];
        colp[0] = (2 * v + pprev + prev) >> 2;

        // Synchronize all threads in a block
        __syncthreads();

        // Loop through each row and apply filter
        for (unsigned row = 2; row < height; row += 2) {
            // Load next element in the column
            uint32_t next = colp[step * (row + 1)];
            uint32_t v = colp[step * row];
            
            // Update current row element
            colp[step * row] = (2 * v + next + prev) >> 2;
            
            // Set current element as previous for next iteration
            prev = next;
        }
    }
}