#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_conv2D_backPropError(double* upStreamError, const double* err, const double* pKernels, size_t kernelCount, size_t kernelRows, size_t kernelCols, size_t outputRows, size_t outputCols, size_t inputChannels, size_t inputRows, size_t inputCols, size_t padding, size_t stride)
{
    // Calculate unique global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    size_t i = id % inputChannels;
    id /= inputChannels;
    size_t j = id % outputRows;
    id /= outputRows;

    if(id >= kernelCount)
        return;

    size_t k = id;

    // Precompute repetitive calculations
    size_t outChannelOffset = k * outputRows * outputCols;
    size_t outRowOffset = j * outputCols;
    int inRowOffset = j * stride - padding;
    size_t kk = k * inputChannels * kernelRows * kernelCols;
    size_t index = outChannelOffset + outRowOffset + i;
    int inColOffset = i * stride - padding;

    // Loop over input channels, kernel rows, and columns
    for(size_t z = 0; z < inputChannels; z++)
    {
        size_t kernelChannelOffset = z * kernelRows * kernelCols;
        size_t inChannelOffset = z * inputRows * inputCols;
        for(size_t y = 0; y < kernelRows; y++)
        {
            size_t kernelRowOffset = y * kernelCols;
            int inRow = inRowOffset + y;
            for(size_t x = 0; x < kernelCols; x++)
            {
                int inCol = inColOffset + x;
                if(inRow >= 0 && inRow < (int)inputRows && inCol >= 0 && inCol < (int)inputCols)
                {
                    size_t idx = inChannelOffset + inputCols * inRow + inCol;
                    atomicAdd(&upStreamError[idx], pKernels[kk + kernelChannelOffset + kernelRowOffset + x] * err[index]);
                    // Using atomicAdd for thread safety 
                }
            }
        }
    }
}