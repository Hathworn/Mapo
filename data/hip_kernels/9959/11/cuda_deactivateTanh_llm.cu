#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_deactivateTanh(double* pE, const double* pA, int n)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use block-level loop unrolling for improved memory access
    #pragma unroll 4
    for (; id < n; id += blockDim.x * gridDim.x) {
        pE[id] *= (1.0 - (pA[id] * pA[id]));
    }
}