#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_deactivateBend(double* pE, const double* pA, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        double x = pE[id];
        // Precompute commonly used term
        double factor = 0.5 / sqrt(x * x + 1);
        // Reduce operations by reusing precomputed term
        pE[id] *= factor * x + 1;
    }
}