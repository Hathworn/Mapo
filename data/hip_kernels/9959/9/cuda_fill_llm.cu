#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_fill(double* pVec, double val, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Each thread writes to its own index, correcting the assignment
    if (id < n)
        pVec[id] = val;
}