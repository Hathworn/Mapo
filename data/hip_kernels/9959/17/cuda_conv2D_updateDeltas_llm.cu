#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_conv2D_updateDeltas(double* delta, double* biasDelta, const double* upStreamActivation, const double* err, double momentum, size_t kernelCount, size_t kernelRows, size_t kernelCols, size_t outputRows, size_t outputCols, size_t inputChannels, size_t inputRows, size_t inputCols, size_t padding, size_t stride)
{
    // Get unique thread id
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    size_t i = id % outputCols;
    id /= outputCols;
    size_t j = id % outputRows;
    id /= outputRows;
    if (id >= kernelCount) return;
    size_t k = id;

    // Precompute offsets to avoid redundant calculations
    size_t outChannelOffset = k * outputRows * outputCols;
    size_t outRowOffset = j * outputCols;
    int inRowOffset = j * stride - padding;
    size_t kk = k * inputChannels * kernelRows * kernelCols;
    size_t index = outChannelOffset + outRowOffset + i;
    int inColOffset = i * stride - padding;

    // Atomic operation to ensure correctness when updating biasDelta
    atomicAdd(&biasDelta[k], err[index]);

    // Loop over input channels and kernel dimensions
    for (size_t z = 0; z < inputChannels; z++) {
        size_t kernelChannelOffset = z * kernelRows * kernelCols;
        size_t inChannelOffset = z * inputRows * inputCols;
        for (size_t y = 0; y < kernelRows; y++) {
            size_t kernelRowOffset = y * kernelCols;
            int inRow = inRowOffset + y;
            for (size_t x = 0; x < kernelCols; x++) {
                int inCol = inColOffset + x;

                // Check boundary conditions for input indices
                if (inRow >= 0 && inRow < (int)inputRows && inCol >= 0 && inCol < (int)inputCols) {
                    size_t idx = inChannelOffset + inputCols * inRow + inCol;

                    // Atomic operation to ensure correctness when updating delta
                    atomicAdd(&delta[kk + kernelChannelOffset + kernelRowOffset + x], err[index] * upStreamActivation[idx]);
                }
            }
        }
    }
}