#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_activateTanh(double* pA, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use alternative calculation of tanh for better performance
    if (id < n) {
        double x = pA[id];
        double e2x = exp(2.0 * x);
        pA[id] = (e2x - 1.0) / (e2x + 1.0); // Utilize exp for tanh calculation
    }
}