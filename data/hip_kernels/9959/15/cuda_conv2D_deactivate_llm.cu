#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_conv2D_deactivate(double* err, const double* net, const double* activation, size_t outputs)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Use __syncwarp() to prevent data hazards and improve memory coalescing
    if(id < outputs) {
        err[id] *= (1.0 - activation[id] * activation[id]);
    }
}