#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_conv2D_ff(double* pA, double* pNet, const double* in, const double* pKernels, const double* pBias, size_t kernelCount, size_t kernelRows, size_t kernelCols, size_t outputRows, size_t outputCols, size_t inputRows, size_t inputCols, size_t inputChannels, size_t padding, size_t stride)
{
    // Use shared memory to improve memory access efficiency
    extern __shared__ double sharedIn[];

    // Calculate thread and block indices
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    size_t i = id % outputCols;
    id /= outputCols;
    size_t j = id % outputRows;
    id /= outputRows;
    if (id >= kernelCount) return;
    size_t k = id;

    // Compute some intermediate values
    size_t outChannelOffset = k * outputRows * outputCols;
    size_t outRowOffset = j * outputCols;
    int inRowOffset = j * stride - padding;
    size_t kk = k * inputChannels * kernelRows * kernelCols;
    size_t index = outChannelOffset + outRowOffset + i;
    int inColOffset = i * stride - padding;

    // Initialize output value with bias
    pNet[index] = pBias[k];

    // Load input data into shared memory
    for (size_t z = 0; z < inputChannels; z++)
    {
        size_t inChannelOffset = z * inputRows * inputCols;
        for (size_t y = 0; y < kernelRows; y++)
        {
            int inRow = inRowOffset + y;
            for (size_t x = 0; x < kernelCols; x++)
            {
                int inCol = inColOffset + x;
                if (inRow >= 0 && inRow < (int)inputRows && inCol >= 0 && inCol < (int)inputCols)
                {
                    size_t idx = inChannelOffset + inputCols * inRow + inCol;
                    sharedIn[threadIdx.x] = in[idx]; // Load input into shared memory
                    __syncthreads();

                    pNet[index] += pKernels[kk + z * kernelRows * kernelCols + y * kernelCols + x] * sharedIn[threadIdx.x];
                    __syncthreads();
                }
            }
        }
    }

    // Apply activation function
    pA[index] = tanh(pNet[index]);
}