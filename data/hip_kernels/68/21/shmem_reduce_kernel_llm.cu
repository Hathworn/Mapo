#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shmem_reduce_kernel(float *d_out, const float *d_in) {
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // Load shared memory from global memory
    sdata[tid] = d_in[myId];
    __syncthreads();  // Ensure entire block is loaded

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();  // Ensure all adds at one stage are done
    }

    // Only thread 0 writes result for this block back to global memory
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}