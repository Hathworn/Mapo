#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in) {
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    // Perform bitonic sort
    for (int stage = 0; stage <= 5; stage++) {
        for (int substage = stage; substage >= 0; substage--) {
            int partner = tid ^ (1 << substage);

            // Ensure partner is within bounds
            if (partner > tid) {
                // Ascending order sort
                if ((tid & (1 << (stage + 1))) == 0) {
                    if (sdata[tid] > sdata[partner]) {
                        // Swap if not in order
                        float temp = sdata[tid];
                        sdata[tid] = sdata[partner];
                        sdata[partner] = temp;
                    }
                } else {
                    // Descending order sort
                    if (sdata[tid] < sdata[partner]) {
                        // Swap if not in order
                        float temp = sdata[tid];
                        sdata[tid] = sdata[partner];
                        sdata[partner] = temp;
                    }
                }
            }
            __syncthreads(); // Sync after each comparison and swap
        }
    }

    d_out[tid] = sdata[tid]; // Write sorted data to output
}