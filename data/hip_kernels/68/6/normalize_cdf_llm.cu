#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n) {
    // Calculate normalization constant once per block to reduce redundant calculations
    __shared__ float normalization_constant;
    if (threadIdx.x == 0) {
        normalization_constant = 1.f / d_input_cdf[n - 1];
    }
    __syncthreads(); // Ensure all threads have the calculated normalization constant

    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unrolling the loop for processing multiple elements per thread
    for(int i = global_index_1d; i < n; i += blockDim.x * gridDim.x) {
        unsigned int input_value = d_input_cdf[i];
        float output_value = input_value * normalization_constant;
        d_output_cdf[i] = output_value;
    }
}