#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void hello() {
    int blockId = blockIdx.x; // Cache block index to reduce register usage
    printf("Hello world! I'm a thread in block %d\n", blockId);
}