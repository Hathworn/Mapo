#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
    // Perform warp-wide reduction using efficient shuffle operations
    unsigned int laneId = threadIdx.x % warpSize;
    
    // Store the input to shared memory
    s[laneId] = p;
    __syncthreads();
    
    // Reduce within the warp using shared memory
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        if (laneId < offset) {
            s[laneId] += s[laneId + offset];
        }
        __syncthreads();
    }
    
    // Return the reduction result
    return s[0];
}

__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in)
{
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    
    // Write reduction result to global memory only if threadIdx.x is 0
    if (t == 0) {
        *d_out_shared = sr;
    }
}