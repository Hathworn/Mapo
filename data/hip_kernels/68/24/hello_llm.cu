#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use shared memory to reduce global memory access
    __shared__ char msg[38];

    // Only the first thread initializes the message
    if (threadIdx.x == 0) {
        strncpy(msg, "Hello world! I'm thread ", 23);
    }

    __syncthreads(); // Ensure message is initialized

    // Use per-thread buffer to store the number
    char buffer[12];
    snprintf(buffer, 12, "%d\n", threadIdx.x);

    // Print message using shared msg and local buffer
    printf("%s%s", msg, buffer);
}