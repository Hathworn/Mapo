#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline int clamp(int value, int min_val, int max_val) {
    return min(max_val, max(min_val, value));
}

__global__ void remove_redness_from_coordinates(const unsigned int* d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, int num_coordinates, int num_pixels_y, int num_pixels_x, int template_half_height, int template_half_width) 
{
    // Calculate global thread index
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;
    int imgSize = num_pixels_x * num_pixels_y;

    // Ensure thread operates only within bounds
    if (global_index_1d < num_coordinates) {
        // Convert 1D coordinate to 2D and adjust
        unsigned int image_index_1d = d_coordinates[imgSize - global_index_1d - 1];
        ushort2 image_index_2d = make_ushort2(image_index_1d % num_pixels_x, image_index_1d / num_pixels_x);

        // Iterate over neighboring pixels
        for (int y = image_index_2d.y - template_half_height; y <= image_index_2d.y + template_half_height; ++y) {
            for (int x = image_index_2d.x - template_half_width; x <= image_index_2d.x + template_half_width; ++x) {
                // Clamp coordinates within valid range
                int clamped_x = clamp(x, 0, num_pixels_x - 1);
                int clamped_y = clamp(y, 0, num_pixels_y - 1);
                int image_offset_index_1d_clamped = (clamped_y * num_pixels_x) + clamped_x;

                // Compute average of green and blue channels
                unsigned char g_value = d_g[image_offset_index_1d_clamped];
                unsigned char b_value = d_b[image_offset_index_1d_clamped];
                unsigned int gb_average = (g_value + b_value) / 2;

                // Update red channel output
                d_r_output[image_offset_index_1d_clamped] = static_cast<unsigned char>(gb_average);
            }
        }
    }
}