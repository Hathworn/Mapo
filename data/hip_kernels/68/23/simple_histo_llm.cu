#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;

    // Avoid accessing out-of-bounds memory
    if (myId >= BIN_COUNT) return;

    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;

    // Minimize atomic operations by using a shared memory buffer
    extern __shared__ int shared_bins[];

    // Initialize shared memory to zero
    if (threadIdx.x < BIN_COUNT) {
        shared_bins[threadIdx.x] = 0;
    }
    __syncthreads();

    // Atomic operation on shared memory
    atomicAdd(&(shared_bins[myBin]), 1);
    __syncthreads();

    // Transfer results from shared memory to global memory
    if (threadIdx.x < BIN_COUNT) {
        atomicAdd(&(d_bins[threadIdx.x]), shared_bins[threadIdx.x]);
    }
}