#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth(float * v_new, const float * v) {
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x; // Correct index calculation
    int numThreads = blockDim.x * gridDim.x;
    if(myIdx < numThreads) { // Ensure thread stays within bounds
        int myLeftIdx = max(myIdx - 1, 0); // Use max to handle left boundary
        int myRightIdx = min(myIdx + 1, numThreads - 1); // Use min to handle right boundary
        float myElt = v[myIdx];
        float myLeftElt = v[myLeftIdx];
        float myRightElt = v[myRightIdx];
        v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt; // Simplified smoothing calculation
    }
}