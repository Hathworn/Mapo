#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void optimized_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Use shared memory to reduce atomic contention
    extern __shared__ int shared_bins[];
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Initialize shared memory
    if (threadIdx.x < BIN_COUNT) {
        shared_bins[threadIdx.x] = 0;
    }
    __syncthreads();

    // Calculate histogram in shared memory
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;
    atomicAdd(&shared_bins[myBin], 1);
    __syncthreads();

    // Write results back to the global histogram
    if (threadIdx.x < BIN_COUNT) {
        atomicAdd(&d_bins[threadIdx.x], shared_bins[threadIdx.x]);
    }
}