#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Kernel01(int N, int M, int P, float *A, float *B, float *C) {
    // Use shared memory to optimize memory access
    __shared__ float sharedA[32][32]; // Assuming block size of 32x32
    __shared__ float sharedB[32][32];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp = 0.0;

    for (int tile = 0; tile < (P + 31) / 32; ++tile) {
        // Load data into shared memory and synchronize
        if (row < N && (tile * 32 + threadIdx.x) < P)
            sharedA[threadIdx.y][threadIdx.x] = A[row * P + tile * 32 + threadIdx.x];
        else
            sharedA[threadIdx.y][threadIdx.x] = 0.0;

        if (col < M && (tile * 32 + threadIdx.y) < P)
            sharedB[threadIdx.y][threadIdx.x] = B[(tile * 32 + threadIdx.y) * M + col];
        else
            sharedB[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        // Perform computation using shared memory
        for (int k = 0; k < 32; ++k) {
            tmp += sharedA[threadIdx.y][k] * sharedB[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < N && col < M) {
        C[row * M + col] = tmp;
    }
}