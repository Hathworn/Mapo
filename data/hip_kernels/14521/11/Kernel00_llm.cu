#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Kernel00(int N, int M, int P, float *A, float *B, float *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) { // Ensure threads remain within bounds
        float tmp = 0.0f;
        for (int k = 0; k < P; k++) {
            tmp += A[row * P + k] * B[k * N + col];
        }
        C[row * N + col] = tmp;
    }
}