#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Kernel11(int N, int M, int P, float *A, float *B, float *C) {

    __shared__ float sA[SIZE][SIZE]; 
    __shared__ float sB[SIZE][SIZE];

    int bx = blockIdx.x; int by = blockIdx.y;
    int tx = threadIdx.x; int ty = threadIdx.y;
    int row = by * SIZE + ty;
    int col = bx * SIZE + tx;

    float tmp = 0.0f;
    int iter = P % SIZE;
    
    // Loop over all blocks of A and B to accumulate results
    for (int m = 0; m < P; m += SIZE) {
        
        // Load elements into shared memory with boundary checks
        if (m + tx < P && row < N) sA[ty][tx] = A[row * P + m + tx];
        else sA[ty][tx] = 0.0f;
        
        if (m + ty < P && col < M) sB[ty][tx] = B[(m + ty) * M + col];
        else sB[ty][tx] = 0.0f;
        
        __syncthreads();
        
        // Perform matrix multiplication on tile
        for (int k = 0; k < SIZE; k++) {
            tmp += sA[ty][k] * sB[k][tx];
        }
        
        __syncthreads();
    }

    // Write result to the output matrix C
    if (row < N && col < M) {
        C[row * M + col] = tmp;
    }
}