#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Kernel01 (int N, int M, int P, float *A, float *B, float *C) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < M) {
        float tmp = 0.0;

        // Use pointer arithmetic to improve memory access patterns
        float* aRow = A + row * P;
        float* bCol = B + col;

        for (int k = 0; k < P; k++) {
            tmp += aRow[k] * bCol[k * N];
        }

        C[row * N + col] = tmp;
    }
}