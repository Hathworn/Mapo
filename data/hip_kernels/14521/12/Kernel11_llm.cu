#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Kernel11(int N, int M, int P, float *A, float *B, float *C) {

// Use shared memory to reduce global memory access
__shared__ float sA[SIZE][SIZE];
__shared__ float sB[SIZE][SIZE];

int bx = blockIdx.x;  
int by = blockIdx.y;
int tx = threadIdx.x; 
int ty = threadIdx.y;
int row = by * SIZE + ty;
int col = bx * SIZE + tx;
int m, k;

// Initialize temporary result
float tmp = 0.0;

// Improve memory coalescing by aligning block iteration
for (m = 0; m < P; m += SIZE) {
    // Load data into shared memory if within bounds
    if (row < N && (m+tx) < P) 
        sA[ty][tx] = A[row * P + m + tx]; 
    else 
        sA[ty][tx] = 0.0;

    if (col < M && (m+ty) < P) 
        sB[ty][tx] = B[(m+ty) * M + col]; 
    else 
        sB[ty][tx] = 0.0;
    __syncthreads();

    // Perform computation for the tile
    for (k = 0; k < SIZE; k++) {
        tmp += sA[ty][k] * sB[k][tx];
    }
    __syncthreads();
}

// Store the result if within the matrix bounds
if ((row < N) && (col < M)) 
    C[row * M + col] = tmp;

}