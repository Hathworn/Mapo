#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZE 16  // Define SIZE as per requirement

__global__ void Kernel11(int N, int M, int P, float *A, float *B, float *C) {

    __shared__ float sA[SIZE][SIZE];
    __shared__ float sB[SIZE][SIZE];

    int bx = blockIdx.x;  
    int by = blockIdx.y;
    int tx = threadIdx.x; 
    int ty = threadIdx.y;
    int row = by * SIZE + ty;
    int col = bx * SIZE + tx;
    float tmp = 0.0;

    for (int m = 0; m < P; m += SIZE) {
        // Load shared memory with boundary check
        sA[ty][tx] = (row < N && (m + tx) < P) ? A[row * P + m + tx] : 0.0f;
        sB[ty][tx] = (col < M && (m + ty) < P) ? B[col + (m + ty) * M] : 0.0f;
        __syncthreads();

        // Perform multiplication
        #pragma unroll  // Unroll loop for better performance
        for (int k = 0; k < SIZE; k++) {
            tmp += sA[ty][k] * sB[k][tx];
        }
        __syncthreads();
    }

    // Store the result in the output matrix C with boundary check
    if ((row < N) && (col < M)) {
        C[row * M + col] = tmp;
    }
}