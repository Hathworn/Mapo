#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Kernel01(int N, int M, int P, float *A, float *B, float *C) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for A and B to optimize memory access
    __shared__ float sharedA[BLOCK_DIM_Y][BLOCK_DIM_X];
    __shared__ float sharedB[BLOCK_DIM_X][BLOCK_DIM_Y];
    
    float tmp = 0.0f;

    // Load data into shared memory
    if (row < N && col < P) {
        sharedA[threadIdx.y][threadIdx.x] = A[row * P + col];
        sharedB[threadIdx.y][threadIdx.x] = B[col * N + row];
    }
    __syncthreads();

    // Compute using shared memory
    if (row < N && col < M) {
        for (int k = 0; k < P; k++) {
            tmp += sharedA[threadIdx.y][k] * sharedB[k][threadIdx.x];
        }
        C[row * N + col] = tmp;
    }
}