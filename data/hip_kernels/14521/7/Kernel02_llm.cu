#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Unroll loop for better performance, use shared memory for A and B
__global__ void Kernel02(int N, int M, int P, float *A, float *B, float *C) {

    __shared__ float shared_A[blockDim.y][blockDim.x];
    __shared__ float shared_B[blockDim.x][blockDim.y];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float tmp = 0.0;
    for (int k = 0; k < P; k += blockDim.x) {
        // Load block of A and B into shared memory
        shared_A[threadIdx.y][threadIdx.x] = (row < N && (k + threadIdx.x) < P) ? A[row * P + k + threadIdx.x] : 0;
        shared_B[threadIdx.x][threadIdx.y] = ((k + threadIdx.y) < P && col < M) ? B[(k + threadIdx.y) * N + col] : 0;
        __syncthreads();
        
        // Unroll loop over block dimension for performance gain
        #pragma unroll
        for (int j = 0; j < blockDim.x; j++) {
            tmp += shared_A[threadIdx.y][j] * shared_B[j][threadIdx.x];
        }
        __syncthreads();
    }
    
    if (row < N && col < M) {
        C[row * N + col] = tmp;
    }
}