#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    extern __shared__ float reduction_buffer[]; // Use dynamic shared memory
    int idx = threadIdx.x;
    int offset = blockIdx.x * dim;

    float sum = 0.0f;

    // Perform first level of reduction
    for (int i = idx; i < dim; i += blockDim.x) {
        sum += dY[offset + i] * Y[offset + i];
    }
    
    reduction_buffer[idx] = sum;
    __syncthreads();

    // Use parallel reduction to accumulate the results
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    // Use the total sum to compute gradient
    float final_sum = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[offset + i] = Y[offset + i] * (dY[offset + i] - final_sum);
    }
}