#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void aggregateEnergies(double *energies, int numEnergies, int interval, int batchSize)
{
    int idx = batchSize * interval * (blockIdx.x * blockDim.x + threadIdx.x);

    // Use shared memory to optimize global memory access
    extern __shared__ double sharedEnergies[];

    // Load data into shared memory
    for (int i = 0; i < batchSize && idx + i * interval < numEnergies; i++) {
        sharedEnergies[i] = energies[idx + i * interval];
    }
    __syncthreads();

    // Reduce values using shared memory
    if (idx < numEnergies) {
        for (int i = 1; i < batchSize; i++) {
            if (idx + i * interval < numEnergies) {
                sharedEnergies[0] += sharedEnergies[i];
            }
        }
        
        // Write back the result to global memory
        energies[idx] = sharedEnergies[0];
        for (int i = 1; i < batchSize; i++) {
            if (idx + i * interval < numEnergies) {
                energies[idx + i * interval] = 0.0;
            }
        }
    }
}