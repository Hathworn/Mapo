#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void aggregateEnergies(double *energies, int numEnergies, int interval, int batchSize)
{
    // Calculate the global thread index
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate initial energy index based on batchSize, interval, and thread index 
    int idx = batchSize * interval * globalIdx;

    // Use shared memory to store energies temporarily for faster access
    extern __shared__ double sharedEnergies[];

    // Load energies into shared memory
    for (int i = 0; i < batchSize; i++) {
        int index = idx + i * interval;
        if (index < numEnergies) {
            sharedEnergies[threadIdx.x * batchSize + i] = energies[index];
        }
    }
    __syncthreads(); // Ensure all threads have loaded their data

    // Aggregate energies in shared memory
    double sum = sharedEnergies[threadIdx.x * batchSize];
    for (int i = 1; i < batchSize; i++) {
        sum += sharedEnergies[threadIdx.x * batchSize + i];
    }
    
    // Store aggregated result back to global memory
    if (idx < numEnergies) {
        energies[idx] = sum; 
    }
    
    // Zero the remaining energies
    for (int i = 1; i < batchSize; i++) {
        int index = idx + i * interval;
        if (index < numEnergies) {
            energies[index] = 0;
        }
    }
}