#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void aggregateEnergies(double *energies, int numEnergies, int interval, int batchSize)
{
    // Calculate global index based on block and thread index
    int idx = batchSize * interval * (blockIdx.x * blockDim.x + threadIdx.x);

    // Check if main index is within bounds
    if (idx < numEnergies) 
    {
        double sum = energies[idx];  // Initialize sum
        for (int i = 1; i < batchSize; i++)
        {
            int offsetIdx = idx + i * interval;
            // Only add if the offset index is within bounds
            if (offsetIdx < numEnergies)
            {
                sum += energies[offsetIdx];
                energies[offsetIdx] = 0;
            }
        }
        energies[idx] = sum;  // Update the main index with the aggregated sum
    }
}