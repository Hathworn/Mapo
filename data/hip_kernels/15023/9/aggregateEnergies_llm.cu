#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void aggregateEnergies(double *energies, int numEnergies, int interval, int batchSize)
{
    int idx = batchSize * interval * (blockIdx.x * blockDim.x + threadIdx.x);
    
    // Early exiting if index is out of bounds
    if (idx >= numEnergies) return;

    // Reduce access to global memory by using a local accumulator
    double accumulator = energies[idx];
    for (int i = 1; i < batchSize; i++)
    {
        int offset = idx + i * interval;
        if (offset < numEnergies)
        {
            accumulator += energies[offset];
            energies[offset] = 0;
        }
    }
    energies[idx] = accumulator;
}