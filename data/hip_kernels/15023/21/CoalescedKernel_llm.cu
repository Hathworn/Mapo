#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void CoalescedKernel(int *x, int *y, int *z, int *sum)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data into shared memory for faster access
    __shared__ int sharedX[256], sharedY[256], sharedZ[256]; // Assuming max blockDim.x is 256
    sharedX[threadIdx.x] = x[idx];
    sharedY[threadIdx.x] = y[idx];
    sharedZ[threadIdx.x] = z[idx];
    __syncthreads();

    // Perform computation using shared memory
    int tempX = sharedX[threadIdx.x];
    int tempY = sharedY[threadIdx.x];
    int tempZ = sharedZ[threadIdx.x];

    sum[idx] = tempX * tempX + tempY * tempY + tempZ * tempZ;
}