#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void calcEnergyParallel(int *atoms, int numAtoms, int *energies, int numEnergies)
{
    int atom1 = blockIdx.x; 
    int atom2 = blockIdx.y * blockDim.x + threadIdx.x;
    if (atom2 >= numAtoms || atom2 <= atom1) return;

    // Calculate energy index more efficiently
    int energyIdx = (2 * numAtoms - atom1 - 3) * atom1 / 2 + atom2 - atom1 - 1;

    energies[energyIdx] = atoms[atom1] * atoms[atom2];
}