#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expandPlanes_kernel_Fp32_NCHW(float* output, const uint64_t* masks, const float* values, int n) {
    // Block size of 256, same mask/val for 64 consecutive threads.
    constexpr int kNumShmemElements = 256 / 64;

    __shared__ uint64_t shMasks[kNumShmemElements];
    __shared__ float shVals[kNumShmemElements];

    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int planeIndex = index >> 6;

    if (planeIndex >= n) return;

    // Improve shared memory access pattern by coalescing reads
    if (threadIdx.x < kNumShmemElements) {
        int localIndex = planeIndex + threadIdx.x;
        shMasks[threadIdx.x] = masks[localIndex];
        shVals[threadIdx.x] = values[localIndex];
    }
    __syncthreads();

    uint64_t mask = shMasks[threadIdx.x >> 6];
    int sqIndex = index & 0x3F;
    float op = 0;

    // Simplify set value check
    if (mask & (1ull << sqIndex)) {
        op = shVals[threadIdx.x >> 6];
    }
    output[index] = op;
}