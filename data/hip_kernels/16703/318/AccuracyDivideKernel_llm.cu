#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel for accuracy division
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory for reduced global memory access
    __shared__ float shared_accuracy;
    
    // Load shared memory from global
    shared_accuracy = *accuracy;
    __syncthreads(); // Ensure all threads have loaded

    // Perform division and write back to global memory
    if (threadIdx.x == 0) {
        shared_accuracy /= N;
        *accuracy = shared_accuracy;
    }
    __syncthreads(); // Synchronize before completion
}