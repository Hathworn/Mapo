#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MxM_naive(double* A, double* B, double* C, const int N) {
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Row i of matrix C
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Column j of matrix C

    if (i < N && j < N) { // Ensure within matrix bounds
        double C_temp = 0.0;
        for (int k = 0; k < N; ++k) { // Compute dot product
            C_temp += A[i * N + k] * B[k * N + j];
        }
        C[i * N + j] = C_temp; // Write output
    }
}