#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_cmp_kernel(std::size_t n, int* aptr, int* bptr, int* rptr) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        // Check if the current element in aptr is less than bptr
        if (aptr[i] < bptr[i]) {
            atomicExch(rptr, 1);  // Use atomic operation to set result pointer
        }
    }
}