#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) { // Ensure within bounds
        c[index] = a[index] + b[index];
    }
}