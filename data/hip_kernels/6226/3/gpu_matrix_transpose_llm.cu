#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_matrix_transpose(int* mat_in, int* mat_out, unsigned int rows, unsigned int cols)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds and perform transpose
    if (idx < cols && idy < rows)
    {
        unsigned int pos = idy * cols + idx;
        unsigned int trans_pos = idx * rows + idy;
        
        // Direct assignment to improve data locality
        mat_out[trans_pos] = mat_in[pos];
    }
}