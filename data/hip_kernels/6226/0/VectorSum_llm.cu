#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t sumWithCuda(int *c, const int *a, const int *b, unsigned int size);

// Global kernel function
__global__ void VectorSum(int *result, const int *vector_a, const int *vector_b)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only perform computation if within bounds
    if (i < size)
    {
        result[i] = vector_a[i] + vector_b[i];
        printf("%d : call kernel function.\n", i);
    }
}