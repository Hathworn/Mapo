#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_square_matrix_mult(int *d_a, int *d_b, int *d_result, int n) 
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub) 
    {
        // Reduce repeated calculations
        int aIdx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        int bIdx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;

        tile_a[threadIdx.y][threadIdx.x] = (aIdx < n * n) ? d_a[aIdx] : 0;
        tile_b[threadIdx.y][threadIdx.x] = (bIdx < n * n) ? d_b[bIdx] : 0;
        
        __syncthreads();

        #pragma unroll  // Suggest unrolling for better performance
        for (int k = 0; k < BLOCK_SIZE; ++k) 
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }

        __syncthreads();
    }
    // Store the result only if within bounds
    if (row < n && col < n) 
    {
        d_result[row * n + col] = tmp;
    }
}