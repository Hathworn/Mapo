#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_matrix_mult(int *a, int *b, int *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    // Check if the thread is within bounds
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            // Optimize access by using more registers efficiently
            int a_val = a[row * n + i];
            int b_val = b[i * k + col];
            sum += a_val * b_val;
        }
        c[row * k + col] = sum;
    }
}