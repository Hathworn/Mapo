#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    __shared__ float uni;
    
    // Load uniform value to shared memory
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];
    
    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x; // Use standard multiplication

    __syncthreads();

    // Add uniform value to two locations in global memory per thread
    if (address < n) {
        g_data[address] += uni;
    }

    if (address + blockDim.x < n) {
        g_data[address + blockDim.x] += uni;
    }
}