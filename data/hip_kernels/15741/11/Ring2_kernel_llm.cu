#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}



#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))


__global__ void Ring2_kernel( float *A, float *BP, int *corrAB, int *mask, int *m, int ring, int c, int h, int w )
{
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;
    if (id1 < size && mask[id1] != 0) {
        int y2 = corrAB[2 * id1 + 1], x2 = corrAB[2 * id1 + 0];
        int x_start = MAX(x2 - ring, 0);
        int x_end = MIN(x2 + ring, w - 1);
        int y_start = MAX(y2 - ring, 0);
        int y_end = MIN(y2 + ring, h - 1);

        for (int dx = x_start; dx <= x_end; dx++)
            for (int dy = y_start; dy <= y_end; dy++)
                m[dy * w + dx] = 1;
    }
}