#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void Ring_kernel(float *A, float *BP, int *corrAB, float *M, int ring, int c, int h, int w) {
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;
    int ringSize = 2 * ring + 1;
    int ringPatch = ringSize * ringSize;

    if (id1 < size) {
        int y1 = id1 / w, x1 = id1 % w;
        int y2 = corrAB[2 * id1 + 1], x2 = corrAB[2 * id1];
        // Use shared memory for the BP patch
        __shared__ float sharedBP[TB];

        for (int dx = -ring; dx <= ring; dx++) {
            for (int dy = -ring; dy <= ring; dy++) {
                int pIdx = (dy + ring) * ringSize + (dx + ring);
                int _x2 = x2 + dx, _y2 = y2 + dy;

                if (_x2 >= 0 && _x2 < w && _y2 >= 0 && _y2 < h) {
                    for (int dc = 0; dc < c; dc++) {
                        // Load BP value into shared memory
                        if (threadIdx.x < TB) {
                            sharedBP[threadIdx.x] = BP[dc * size + _y2 * w + _x2];
                            __syncthreads();
                        }
                        // Use shared memory to fill M
                        M[(dc * size + y1 * w) * ringPatch + pIdx * w + x1] = sharedBP[threadIdx.x];
                    }
                }
            }
        }
    }
}