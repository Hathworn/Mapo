#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void patchmatch_r_argmax_kernel(float *conv, float *target, float *match, int *correspondence, int c1, int h1, int w1, int h2, int w2) {
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int size1 = h1 * w1, size2 = h2 * w2;

    if (id1 < size1) {
        // Precalculate indices and reduce redundant calculations
        int baseIdx = id1 * size2;
        int corrBaseIdx = id1 * 2;

        double conv_max = -1e20;
        int best_x2 = 0, best_y2 = 0;

        for (int id2 = 0; id2 < size2; id2++) {
            float conv_result = conv[baseIdx + id2];
            if (conv_result > conv_max) {
                conv_max = conv_result;
                best_x2 = id2 % w2;
                best_y2 = id2 / w2;
            }
        }

        correspondence[corrBaseIdx + 0] = best_x2;
        correspondence[corrBaseIdx + 1] = best_y2;

        // Reduce calculation inside the loop
        for (int c = 0; c < c1; c++) {
            match[c * size1 + id1] = target[c * size2 + best_y2 * w2 + best_x2];
        }
    }
}