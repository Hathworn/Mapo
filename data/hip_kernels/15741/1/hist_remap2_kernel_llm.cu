#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void hist_remap2_kernel( float *I, int nI, float *mI, float *histJ, float *cumJ, float *_minJ, float *_maxJ, int nbins, float *_sortI, int *_idxI, float *R, int c, int h, int w )
{
    int _id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;

    if (_id < c * size) {
        // Calculate once outside the loop
        int id = _id % size, dc = _id / size;

        float minJ = _minJ[dc];
        float maxJ = _maxJ[dc];
        float stepJ = (maxJ - minJ) / nbins;

        int idxI = _idxI[_id] - 1;
        if (mI[idxI] < EPS)
            return;
        int offset = h * w - nI;
        int cdf = id - offset;

        int s = 0, e = nbins - 1, m;
        int binIdx = -1;

        // Use binary search efficiently
        while (s <= e) {
            m = (s + e) >> 1; // Use bitwise shift for division by 2
            float cdf_e = (m == nbins - 1) ? cumJ[dc * nbins + m] + 0.5f : cumJ[dc * nbins + m];
            float cdf_s = (m == 0) ? -0.5f : cumJ[dc * nbins + m - 1];

            if (cdf >= cdf_e) {
                s = m + 1;
            } else if (cdf < cdf_s) {
                e = m - 1;
            } else {
                binIdx = m;
                break;
            }
        }

        if (binIdx >= 0) { // Ensure valid binIdx before accessing arrays
            float hist = histJ[dc * nbins + binIdx];
            float cdf_e = cumJ[dc * nbins + binIdx];
            float cdf_s = cdf_e - hist;
            float ratio = MIN(MAX((cdf - cdf_s) / (hist + 1e-8), 0.0f), 1.0f);
            float activation = minJ + (static_cast<float>(binIdx) + ratio) * stepJ;
            R[dc * size + idxI] = activation;
        }
    }
}