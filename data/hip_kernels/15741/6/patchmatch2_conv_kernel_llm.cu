#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void patchmatch2_conv_kernel( float *A, float *B, float *AP, float *BP, float *conv, int *prev_corrAB_upsampled, int patch, int s_rad, int c, int h, int w )
{
    int _id = blockIdx.x * blockDim.x + threadIdx.x;
    int size1 = h * w;
    int s_size = 2 * s_rad + 1;
    int s_n = s_size * s_size;

    if (_id < size1 * s_n) {
        conv[_id] = -1;

        int id1 = _id / s_n, s_idx = _id % s_n;
        int y1 = id1 / w, x1 = id1 % w;
        int dy2 = s_idx / s_size - s_rad, dx2 = s_idx % s_size - s_rad;

        int x2 = prev_corrAB_upsampled[2 * id1 + 0];
        int y2 = prev_corrAB_upsampled[2 * id1 + 1];

        int new_y2 = y2 + dy2;
        int new_x2 = x2 + dx2;
        if (!(new_x2 >= 0 && new_x2 < w && new_y2 >= 0 && new_y2 < h)) {
            return;
        }

        // Utilize shared memory for accessing data in a block
        __shared__ float sharedA[TB];
        __shared__ float sharedB[TB];
        __shared__ float sharedAP[TB];
        __shared__ float sharedBP[TB];

        int kernel_radius = (patch - 1) / 2;
        float conv_result = 0;
        int cnt = 0;

        for (int dy = -kernel_radius; dy <= kernel_radius; dy++) {
            for (int dx = -kernel_radius; dx <= kernel_radius; dx++) {
                int xx1 = x1 + dx, yy1 = y1 + dy;
                int xx2 = new_x2 + dx, yy2 = new_y2 + dy;
                if (0 <= xx1 && xx1 < w && 0 <= yy1 && yy1 < h &&
                    0 <= xx2 && xx2 < w && 0 <= yy2 && yy2 < h)
                {
                    int _id1 = yy1 * w + xx1, _id2 = yy2 * w + xx2;
                    for (int dc = 0; dc < c; dc++) {
                        // Assign shared memory variables
                        int sharedIdx = threadIdx.x;
                        sharedA[sharedIdx] = A[dc * size1 + _id1];
                        sharedB[sharedIdx] = B[dc * size1 + _id2];
                        sharedAP[sharedIdx] = AP[dc * size1 + _id1];
                        sharedBP[sharedIdx] = BP[dc * size1 + _id2];

                        __syncthreads(); // Ensure all threads have loaded data

                        conv_result += sharedA[sharedIdx] * sharedB[sharedIdx];
                        conv_result += sharedAP[sharedIdx] * sharedBP[sharedIdx];
                    }
                    cnt++;
                }
            }
        }

        if (cnt > 0)
            conv[_id] = conv_result / cnt;
    }
    return;
}