#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void patchmatch2_argmax_kernel(float *conv, int *prev_corrAB_upsampled, int *corrAB, int s_rad, int c, int h, int w) {
    int h1 = h, w1 = w;
    int id1 = blockIdx.x * blockDim.x + threadIdx.x;
    int size1 = h1 * w1;
    int s_size = 2 * s_rad + 1;
    int s_n = s_size * s_size;

    if (id1 < size1) {
        float conv_max = -1;

        int x2 = prev_corrAB_upsampled[2 * id1 + 0];
        int y2 = prev_corrAB_upsampled[2 * id1 + 1];

        int max_x = max(0, x2 - s_rad);
        int max_y = max(0, y2 - s_rad);
        int min_x = min(w1 - 1, x2 + s_rad);
        int min_y = min(h1 - 1, y2 + s_rad);

        for (int dx2 = max_x - x2, dx2_end = min_x - x2; dx2 <= dx2_end; dx2++) {
            for (int dy2 = max_y - y2, dy2_end = min_y - y2; dy2 <= dy2_end; dy2++) {
                int new_x2 = x2 + dx2;
                int new_y2 = y2 + dy2;

                int s_idx = (dy2 + s_rad) * s_size + (dx2 + s_rad);
                int id = id1 * s_n + s_idx;
                float conv_result = conv[id];
                if (conv_result > conv_max) {
                    conv_max = conv_result;
                    corrAB[id1 * 2 + 0] = new_x2;
                    corrAB[id1 * 2 + 1] = new_y2;
                }
            }
        }
    }
}