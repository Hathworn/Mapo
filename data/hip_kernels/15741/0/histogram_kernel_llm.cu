#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void histogram_kernel(float *I, float *minI, float *maxI, float *mask, int nbins, int c, int h, int w, float *hist) {
    int _id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;

    if (_id >= c * size) return;  // Exit early if outside bounds

    int id = _id % size, dc = _id / size;

    if (mask[id] < EPS) return;  // Skip processing if mask condition is not met

    float val = I[_id];
    float _minI = minI[dc];
    float _maxI = maxI[dc];

    if (_minI == _maxI) {  // Adjust bounds if they are equal
        _minI -= 1;
        _maxI += 1;
    }

    if (_minI <= val && val <= _maxI) {  // Only process values within range
        int idx = MIN((val - _minI) / (_maxI - _minI) * nbins, nbins - 1);
        int index = dc * nbins + idx;
        atomicAdd(&hist[index], 1.0f);  // Safely update the histogram
    }
}