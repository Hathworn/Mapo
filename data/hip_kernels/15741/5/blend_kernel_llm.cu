#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 0.1

#undef MIN
#define MIN(a, b) ((a) < (b) ? (a) : (b))

#undef MAX
#define MAX(a, b) ((a) > (b) ? (a) : (b))

__global__ void blend_kernel(float *A, float *BP, float *M, float *AP, float alpha, int c, int h, int w)
{
    // Calculate unique thread ID
    int _id = blockIdx.x * blockDim.x + threadIdx.x;
    // Precompute the total size
    int size = h * w;

    // Ensure calculations only happen within valid bounds
    if (_id < c * size) {
        // Efficiently calculate id and dc
        int id = _id % size;
        int dc = _id / size;

        // Simplify weight calculation using ternary operator
        float weight = (M[id] < 0.05f) ? 0.f : alpha;

        // Direct computation for output assignment
        AP[_id] = A[_id] * weight + BP[_id] * (1.f - weight);
    }
}