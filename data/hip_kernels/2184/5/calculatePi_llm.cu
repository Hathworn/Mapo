#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function
__global__ void calculatePi(double *piTotal, long int iterations, int totalThreads)
{
    long int i;
    double piPartial = 0.0;

    int index = (blockDim.x * blockIdx.x) + threadIdx.x;
    long int initIteration = (iterations / totalThreads) * index;
    long int endIteration = initIteration + (iterations / totalThreads);

    // Unrolling loop for improved performance
    for (i = initIteration; i < endIteration; i++)
    {
        piPartial += (4.0 / (i * 2 + 1)) - (4.0 / (i * 2 + 3));
    }

    piTotal[index] = piPartial;

    __shared__ double sharedPiTotal[1024];  // Use shared memory to reduce global memory access
    sharedPiTotal[threadIdx.x] = piPartial;

    __syncthreads();

    // Reduction to sum results across threads using shared memory
    if (threadIdx.x == 0)
    {
        double blockSum = 0.0;
        for (int j = 0; j < blockDim.x; j++)
        {
            blockSum += sharedPiTotal[j];
        }
        atomicAdd(&piTotal[0], blockSum);  // Use atomicAdd to safely accumulate final result
    }
}