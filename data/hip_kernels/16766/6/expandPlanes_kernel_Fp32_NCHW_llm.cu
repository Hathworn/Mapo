#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expandPlanes_kernel_Fp32_NCHW(float* output, const uint64_t* masks, const float* values, int n) {
    // Improved block and shared memory access
    constexpr int kNumShmemElements = 256 / 64;

    __shared__ uint64_t shMasks[kNumShmemElements];
    __shared__ float shVals[kNumShmemElements];

    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int laneId = threadIdx.x & 0x3F; // Use lane ID for mask/index

    int planeIndex = index >> 6;

    if (planeIndex >= n) return;

    // Load masks and values into shared memory for the whole warp
    if(laneId == 0) {
        shMasks[threadIdx.x >> 6] = masks[planeIndex];
        shVals[threadIdx.x >> 6] = values[planeIndex];
    }
    __syncthreads();

    // Broadcast data from shared memory
    uint64_t mask = shMasks[threadIdx.x >> 6];
    float op = (mask & (1ull << laneId)) ? shVals[threadIdx.x >> 6] : 0;

    output[index] = op;
}