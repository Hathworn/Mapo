#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load the accuracy into a register for faster access
    float acc = *accuracy;
    
    // Perform division in the register
    acc /= N;
    
    // Store the result back to memory
    *accuracy = acc;
}