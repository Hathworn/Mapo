#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void double_value(double *x, double *y)
{
    // Compute global thread index for flexible loop handling and coalescing
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the global index does not exceed the array bounds
    if (idx < N)
    {
        y[idx] = 2.0 * x[idx];
    }
}