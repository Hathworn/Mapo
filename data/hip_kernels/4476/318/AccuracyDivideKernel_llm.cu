#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use a cleaner operation by reducing atomic operation and handling edge case
    if (N != 0) {
        *accuracy = __fdividef(*accuracy, N); // Use faster float division intrinsic
    }
}