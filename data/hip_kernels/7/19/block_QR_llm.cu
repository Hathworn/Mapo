#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void block_QR(float* z, float* z1, float* vector, float* vector1, float* Q, float* NewQ, float* R, float* PrevM, float* NewM, int* converged, float* eigenvector, const int *WidthOfMatrix, const int *ind, const int *vind)
{
    // Shared memory for intermediate computations
    extern __shared__ float sharedMem[];

    int n = WidthOfMatrix[blockIdx.x];
    int index = ind[blockIdx.x];
    int vectindex = vind[blockIdx.x];
    int numofelements = n*n;

    // Initialize convergence flag
    if(threadIdx.x == 0) {
        converged[blockIdx.x] = 0;
    }
    
    // Initialize matrices and eigenvectors
    for(int i = threadIdx.x; i < numofelements; i += blockDim.x) {
        int iplusindex = i + index;
        eigenvector[iplusindex] = (i / n == i % n) ? 1.0f : 0.0f;
        z1[iplusindex] = z[iplusindex];
        Q[iplusindex] = z[iplusindex];
        PrevM[iplusindex] = z[iplusindex];
    }
    __syncthreads();

    // Main QR decomposition loop
    do {
        for(int k = 0; k < n - 1; k++) {
            float NormCheck = z[k*n+k+index];

            // Step 1: Set minor matrix
            for(int i = threadIdx.x; i < numofelements; i += blockDim.x) {
                int row = i / n, col = i % n;
                z[i+index] = (row == col && row < k) ? 1.0f : (row >= k && col >= k) ? z[i+index] : 0.0f;
            }
            __syncthreads();

            // Step 2: Extract k-th column
            for(int i = threadIdx.x; i < n; i += blockDim.x) {
                vector[i+vectindex] = z[i*n+k+index];
            }
            __syncthreads();

            // Step 3: Compute norm
            float NormOfKcol = 0.0f;
            for(int i = threadIdx.x; i < n; i += blockDim.x) {
                NormOfKcol += vector[i+vectindex] * vector[i+vectindex];
            }
            NormOfKcol = sqrt(NormOfKcol);
            if(NormCheck > 0) NormOfKcol = -NormOfKcol;
            if(k == threadIdx.x) vector[k+vectindex] += NormOfKcol;
            __syncthreads();

            // Step 8: Normalize vector
            for(int i = threadIdx.x; i < n; i += blockDim.x) {
                vector[i+vectindex] /= NormOfKcol;
            }
            __syncthreads();

            // Step 9: Compute Vmul
            for(int i = threadIdx.x; i < numofelements; i += blockDim.x) {
                R[i+index] = -2 * vector[i/n+vectindex] * vector[i%n+vectindex];
                if(i/n == i%n) R[i+index] += 1.0f;
            }
            __syncthreads();

            // Step 10: Multiply Vmul by z1
            for(int i = threadIdx.x; i < numofelements; i += blockDim.x) {
                float sum = 0.0f;
                for(int j = 0; j < n; j++) {
                    sum += R[i/n*n+j+index] * z1[j*n+i%n+index];
                }
                z[i+index] = sum;
            }
            __syncthreads();

            // Step 11 & 12: Update Q
            if(k != 0) {
                for(int i = threadIdx.x; i < numofelements; i += blockDim.x) {
                    float sum = 0.0f;
                    for(int j = 0; j < n; j++) {
                        sum += R[i/n*n+j+index] * Q[j*n+i%n+index];
                    }
                    NewQ[i+index] = sum;
                }
                __syncthreads();
                for(int i = threadIdx.x; i < numofelements; i += blockDim.x) {
                    Q[i+index] = NewQ[i+index];
                }
            } else {
                for(int i = threadIdx.x; i < numofelements; i += blockDim.x) {
                    Q[i+index] = R[i+index];
                }
            }
            __syncthreads();
            
            for(int i = threadIdx.x; i < numofelements; i += blockDim.x) {
                z1[i+index] = z[i+index];
            }
            __syncthreads();
        }

        // Step 13: Compute R
        for(int i = threadIdx.x; i < numofelements; i += blockDim.x) {
            R[i+index] = 0.0f;
            for(int j = 0; j < n; j++) {
                R[i+index] += Q[i/n*n+j+index] * PrevM[j*n+i%n+index];
            }
        }
        __syncthreads();

        // Step 14: Transpose Q
        for(int i = threadIdx.x; i < numofelements; i += blockDim.x) {
            z[i%n*n + i/n + index] = Q[i+index];
        }
        __syncthreads();

        // Step 14.5: Update eigenvectors
        for(int i = threadIdx.x; i < numofelements; i += blockDim.x) {
            float sum = 0.0f;
            for(int j = 0; j < n; j++) {
                sum += eigenvector[i/n*n + j + index] * z[j*n + i%n + index];
            }
            NewM[i+index] = sum;
        }
        __syncthreads();
        for(int i = threadIdx.x; i < numofelements; i += blockDim.x) {
            eigenvector[i+index] = NewM[i+index];
        }
        __syncthreads();

        // Step 15: Multiply R and TransposeOfQ
        for(int i = threadIdx.x; i < numofelements; i += blockDim.x) {
            float sum = 0.0f;
            for(int j = 0; j < n; j++) {
                sum += R[i/n*n+j+index] * z[j*n+i%n+index];
            }
            NewM[i+index] = sum;
        }
        __syncthreads();

        // Step 16: Check convergence
        if(threadIdx.x == 0) {
            converged[blockIdx.x] = 1;
        }
        __syncthreads();
        
        for(int i = threadIdx.x; i < numofelements; i += blockDim.x) {
            if(i/n == i%n && (PrevM[i+index]/NewM[i+index] > 1.000001f || PrevM[i+index]/NewM[i+index] < 0.999999f)) {
                converged[blockIdx.x] = 0;
            }
        }
        __syncthreads();

        // Step 17: Prepare for next iteration if not converged
        if(converged[blockIdx.x] == 0) {
            for(int i = threadIdx.x; i < numofelements; i += blockDim.x) {
                int iplusindex = i + index;
                z[iplusindex] = NewM[iplusindex];
                z1[iplusindex] = NewM[iplusindex];
                Q[iplusindex] = NewM[iplusindex];
                PrevM[iplusindex] = NewM[iplusindex];
            }
        }
        __syncthreads();
    } while(converged[blockIdx.x] == 0);

    // Store eigenvalues
    if(threadIdx.x < n) {
        vector[threadIdx.x+vectindex] = NewM[threadIdx.x+threadIdx.x*n+index];
    }
    __syncthreads();

    // Sort eigenvalues and corresponding eigenvectors
    if(threadIdx.x == 0) {
        for(int i1 = vectindex; i1 < n - 1 + vectindex; i1++) {
            for(int i2 = i1 + 1; i2 < n + vectindex; i2++) {
                if(vector[i1] > vector[i2]) {
                    float tmpVal = vector[i1];
                    vector[i1] = vector[i2];
                    vector[i2] = tmpVal;
                    
                    for(int i3 = 0; i3 < n; i3++) {
                        float tmpVec = eigenvector[i3*n + (i1-vectindex)%n + index];
                        eigenvector[i3*n + (i1-vectindex)%n + index] = eigenvector[i3*n + (i2-vectindex)%n + index];
                        eigenvector[i3*n + (i2-vectindex)%n + index] = tmpVec;
                    }
                }
            }
        }
    }
}