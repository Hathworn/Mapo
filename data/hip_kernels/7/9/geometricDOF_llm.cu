#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void geometricDOF(float *Qi_gdof, float4 *positions, float *masses, int *blocknums, int *blocksizes, int largestsize, float *norm, float *pos_center) {
    int blockNum = blockIdx.x * blockDim.x + threadIdx.x;

    // Pre-compute offsets and reduce calculation inside the loop
    int baseIdx = blockNum * largestsize * 6;
    int blockSize = blocksizes[blockNum];
    int blockStart = blocknums[blockNum];

    // Loop through atoms in blocks
    for (int j = 0; j < blockSize - 3; j += 3) {
        int atom = (blockStart + j) / 3;
        float mass = masses[atom];
        float factor = sqrt(mass) / norm[atom];

        // Fill the diagonal
        Qi_gdof[baseIdx + j * 6 + 0] = factor;
        Qi_gdof[baseIdx + (j + 1) * 6 + 1] = factor;
        Qi_gdof[baseIdx + (j + 2) * 6 + 2] = factor;

        // Compute differences
        float diff0 = positions[atom].x - pos_center[atom * 3 + 0];
        float diff1 = positions[atom].y - pos_center[atom * 3 + 1];
        float diff2 = positions[atom].z - pos_center[atom * 3 + 2];

        // Fill the off-diagonal
        Qi_gdof[baseIdx + (j + 1) * 6 + 3] = diff2 * factor;
        Qi_gdof[baseIdx + (j + 2) * 6 + 3] = -diff1 * factor;

        Qi_gdof[baseIdx + j * 6 + 4] = -diff2 * factor;
        Qi_gdof[baseIdx + (j + 2) * 6 + 4] = -diff0 * factor;

        Qi_gdof[baseIdx + j * 6 + 5] = diff1 * factor;
        Qi_gdof[baseIdx + (j + 1) * 6 + 5] = -diff0 * factor;
    }

    // Normalize first vector
    float rotnorm = 0.0f;
    for (int j = 0; j < blockSize; j++) {
        rotnorm += Qi_gdof[baseIdx + j * 6 + 3] * Qi_gdof[baseIdx + j * 6 + 3];
    }

    rotnorm = rsqrtf(rotnorm); // Use fast inverse square root for normalization

    for (int j = 0; j < blockSize; j++) {
        Qi_gdof[baseIdx + j * 6 + 3] *= rotnorm;
    }
}