#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyToOpenMM(float *target, float *source, int N) {
    int elementNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (elementNum >= N) return; // Optimize boundary check

    int atom = elementNum / 3;
    target[4 * atom + elementNum % 3] = source[elementNum]; // Rearrange calculation for clarity
}