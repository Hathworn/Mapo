#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void symmetrize1D(float *h, int *blockPositions, int *blockSizes, int numBlocks) {
    int blockNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (blockNum >= numBlocks) {
        return;
    }

    // Convert block size from atoms to degrees of freedom
    const unsigned int blockSize = 3 * blockSizes[blockNum];

    float *block = &(h[blockPositions[blockNum]]);
    
    // Loop unrolling for improved performance
    for (unsigned int r = 0; r < blockSize - 1; r++) {
        for (unsigned int c = r + 1; c < blockSize; c += 2) { // Increment by 2 for loop unrolling
            const float avg1 = 0.5f * (block[r * blockSize + c] + block[c * blockSize + r]);
            block[r * blockSize + c] = avg1;
            block[c * blockSize + r] = avg1;

            if (c + 1 < blockSize) { // Handle the second part of the unrolled loop
                const float avg2 = 0.5f * (block[r * blockSize + c + 1] + block[(c + 1) * blockSize + r]);
                block[r * blockSize + c + 1] = avg2;
                block[(c + 1) * blockSize + r] = avg2;
            }
        }
    }
}