#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void makeEigenvalues(float *eigenvalues, float *blockHessian, int *blocknums, int *blocksizes, int *hessiannums, int N, int numblocks) {
    // Use shared memory to reduce repetitive calculations and increase efficiency
    __shared__ int sharedBlockNums[1024];
    __shared__ int sharedHessianNums[1024];
    __shared__ int sharedBlockSizes[1024];

    int elementNum = blockIdx.x * blockDim.x + threadIdx.x;

    if (elementNum >= N) {
        return;
    }

    if(threadIdx.x < numblocks) {
        sharedBlockNums[threadIdx.x] = blocknums[threadIdx.x];
        sharedHessianNums[threadIdx.x] = hessiannums[threadIdx.x];
        sharedBlockSizes[threadIdx.x] = blocksizes[threadIdx.x];
    }
    __syncthreads();

    int b = 0;

    // Optimize: Unroll the loop and use shared memory
    for (int i = 0; i < numblocks; i++) {
        if (sharedBlockNums[i] > elementNum / 3) {
            b = i - 1;
            break;
        }
    }

    int x = elementNum - 3 * sharedBlockNums[b];
    
    // Calculate the offset using the block's starting position and size
    int spot = sharedHessianNums[b] + x * (3 * sharedBlockSizes[b]) + x;

    eigenvalues[elementNum] = blockHessian[spot];
}