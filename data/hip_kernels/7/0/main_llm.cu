#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "makeHE_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *HE = NULL;
hipMalloc(&HE, XSIZE*YSIZE);
float *force1 = NULL;
hipMalloc(&force1, XSIZE*YSIZE);
float4 *force2 = NULL;
hipMalloc(&force2, XSIZE*YSIZE);
float *masses = NULL;
hipMalloc(&masses, XSIZE*YSIZE);
float eps = 1;
int k = 1;
int m = 2;
int N = XSIZE*YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
makeHE<<<gridBlock, threadBlock>>>(HE,force1,force2,masses,eps,k,m,N);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
makeHE<<<gridBlock, threadBlock>>>(HE,force1,force2,masses,eps,k,m,N);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
makeHE<<<gridBlock, threadBlock>>>(HE,force1,force2,masses,eps,k,m,N);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}