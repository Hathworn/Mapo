#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void makeHE( float *HE, float *force1, float4 *force2, float *masses, float eps, int k, int m, int N ) {
    int elementNum = blockIdx.x * blockDim.x + threadIdx.x;
    if(elementNum >= N) return; // Early return if out of bounds
    
    int atom = elementNum / 3;
    int axis = elementNum % 3;
    
    // Use an array to simplify indexing and eliminate branching
    float force_difference[3] = {force2[atom].x, force2[atom].y, force2[atom].z}; 
    HE[elementNum * m + k] = (force1[elementNum] - force_difference[axis]) / (sqrt(masses[atom]) * eps);
}