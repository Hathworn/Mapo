#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void symmetrize2D(float *h, int natoms) {
    const int elementNum = blockIdx.x * blockDim.x + threadIdx.x;
    const int dof = 3 * natoms;

    if (elementNum < dof * dof) { // Process only valid elements
        int r = elementNum / dof;
        int c = elementNum % dof;

        if (r <= c) { // Only handle the upper triangular and diagonal
            const float avg = 0.5f * (h[r * dof + c] + h[c * dof + r]);
            h[r * dof + c] = avg;
            h[c * dof + r] = avg;
        }
    }
}