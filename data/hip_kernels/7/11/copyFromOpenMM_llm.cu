#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyFromOpenMM(float *target, float *source, int N) {
    const int elementNum = blockIdx.x * blockDim.x + threadIdx.x;
    // Boundary check adjusted to be less-than to handle exact boundary cases
    if (elementNum < N) {
        const int atom = elementNum / 3;
        // Streamline the access to avoid repeated calculations
        target[elementNum] = source[atom * 4 + elementNum % 3];
    }
}