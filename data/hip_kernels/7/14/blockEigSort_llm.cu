#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blockEigSort( float *eigenvalues, float *eigenvectors, int *blocknums, int *blocksizes, int N ) {
    int blockNumber = blockIdx.x * blockDim.x + threadIdx.x;
    int startspot = blocknums[blockNumber];
    int endspot = startspot + blocksizes[blockNumber] - 1;

    // Use a more efficient sorting algorithm: selection sort
    for(int i = startspot; i < endspot; i++) {
        int minIndex = i;
        for(int j = i + 1; j <= endspot; j++) {
            if(eigenvalues[j] < eigenvalues[minIndex]) {
                minIndex = j; 
            }
        }
        if(minIndex != i) {
            float tmp = eigenvalues[i];
            eigenvalues[i] = eigenvalues[minIndex];
            eigenvalues[minIndex] = tmp;

            // Swap associated eigenvectors
            for(int k = 0; k < N; k++) {
                tmp = eigenvectors[k * N + i];
                eigenvectors[k * N + i] = eigenvectors[k * N + minIndex];
                eigenvectors[k * N + minIndex] = tmp;
            }
        }
    }
}