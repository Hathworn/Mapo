#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatMulKernel( float *C, float *A, float *B, int Aheight, int Awidth, int Bwidth ) {
    // Compute elementNum based on indices
    int elementNum = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we do not compute out of bounds
    if( elementNum >= Aheight * Bwidth ) {
        return;
    }
    
    // Calculate row and column index
    int row = elementNum / Bwidth;
    int col = elementNum % Bwidth;
    
    float result = 0; // Initialize result accumulator

    // Iterate over Awidth to perform matrix multiplication
    for( int e = 0; e < Awidth; e++ ) {
        result += A[row * Awidth + e] * B[e * Bwidth + col]; // Accumulate product
    }
    
    // Store result in C
    C[row * Bwidth + col] = result;
}