#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "computeNormsAndCenter.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *norms = NULL;
hipMalloc(&norms, XSIZE*YSIZE);
float *center = NULL;
hipMalloc(&center, XSIZE*YSIZE);
float *masses = NULL;
hipMalloc(&masses, XSIZE*YSIZE);
float4 *positions = NULL;
hipMalloc(&positions, XSIZE*YSIZE);
int *blocknums = NULL;
hipMalloc(&blocknums, XSIZE*YSIZE);
int *blocksizes = NULL;
hipMalloc(&blocksizes, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
computeNormsAndCenter<<<gridBlock, threadBlock>>>(norms,center,masses,positions,blocknums,blocksizes);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
computeNormsAndCenter<<<gridBlock, threadBlock>>>(norms,center,masses,positions,blocknums,blocksizes);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
computeNormsAndCenter<<<gridBlock, threadBlock>>>(norms,center,masses,positions,blocknums,blocksizes);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}