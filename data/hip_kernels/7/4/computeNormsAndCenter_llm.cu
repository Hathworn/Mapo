#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeNormsAndCenter( float *norms, float *center, float *masses, float4 *positions, int *blocknums, int *blocksizes ) {
    int blockNum = blockIdx.x * blockDim.x + threadIdx.x;
    float3 cent = make_float3(0.0f, 0.0f, 0.0f); // Use float3 for center calculation
    float totalmass = 0.0f;
    
    // Loop unrolling for potential increase in performance
    int start = blocknums[blockNum];
    int end = blocknums[blockNum] + blocksizes[blockNum] - 1;
    for( int j = start; j <= end; j += 3 ) {
        int index = j / 3;
        float mass = masses[index];
        float4 pos = positions[index];
        
        cent.x += pos.x * mass;
        cent.y += pos.y * mass;
        cent.z += pos.z * mass;
        totalmass += mass;
    }

    norms[blockNum] = sqrt(totalmass);
    center[blockNum * 3 + 0] = cent.x / totalmass;
    center[blockNum * 3 + 1] = cent.y / totalmass;
    center[blockNum * 3 + 2] = cent.z / totalmass;
}