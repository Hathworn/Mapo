#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blockcopyFromOpenMM(float *target, float *source, int *blocks, int numblocks, int setnum, int N) {
    int blockNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (blockNum >= numblocks)
        return; // Out of bounds check for blockNum
    
    int dof = 3 * blocks[blockNum] + setnum;
    int atom = dof / 3;

    if(atom >= N || (blockNum != numblocks - 1 && atom >= blocks[blockNum + 1])) {
        return; // Out of bounds
    }

    target[dof] = source[(dof + atom + 1)]; // Directly use source index without calculating with sizeof
}