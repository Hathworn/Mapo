#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void makeProjection(float *eT, float *e, float *eigenvec, int *indices, int M, int N) {
    // Calculate the indices only once
    int elementNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (elementNum >= M * N) {
        return;
    }
    
    int m = elementNum / N;
    int n = elementNum % N;

    // Calculate global index for reuse
    int globalIndex = n * M + m;
    int eigenvecIndex = n * M + indices[m];

    // Use globalIndex for setting both e and eT
    float val = eigenvec[eigenvecIndex];
    e[globalIndex] = val;
    eT[m * N + n] = val;
}