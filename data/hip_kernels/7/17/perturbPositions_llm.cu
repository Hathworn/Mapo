#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void perturbPositions(float *blockpos, float4 *initpos, float delta, int *blocks, int numblocks, int setnum, int N) {
    int blockNum = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (blockNum >= numblocks) {
        return;
    }
    
    int dof = 3 * blocks[blockNum] + setnum;
    int atom = dof / 3;

    // Ensure atom is within bounds
    if (atom >= N || (blockNum != (numblocks - 1) && atom >= blocks[blockNum + 1])) {
        return;
    }

    int axis = dof % 3;

    // Use a switch statement for cleaner code
    switch (axis) {
        case 0:
            blockpos[dof] = initpos[atom].x;
            initpos[atom].x += delta;
            break;
        case 1:
            blockpos[dof] = initpos[atom].y;
            initpos[atom].y += delta;
            break;
        case 2:
            blockpos[dof] = initpos[atom].z;
            initpos[atom].z += delta;
            break;
    }
}