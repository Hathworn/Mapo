#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blockcopyToOpenMM(float *target, float *source, int *blocks, int numblocks, int setnum, int N) {
    int blockNum = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if blockNum is out of bounds
    if (blockNum >= numblocks)
        return;

    int dof = 3 * blocks[blockNum] + setnum;
    int atom = dof / 3;

    // Combined condition to check if atom is out of bounds
    if (atom >= N || (blockNum + 1 < numblocks && atom >= blocks[blockNum + 1]))
        return; // Out of bounds

    // Efficient memory access pattern
    target[(dof + atom + 1)] = source[dof];
}