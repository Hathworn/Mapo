#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void perturbByE(float *tmppos, float4 *mypos, float eps, float *E, float *masses, int k, int m, int N) {
    int dof = blockIdx.x * blockDim.x + threadIdx.x;
    if (dof >= N) return;

    int atom = dof / 3;
    int axis = dof % 3;

    // Precompute common terms
    float scaleFactor = eps * E[dof * m + k] / sqrtf(masses[atom]);

    // Use shared memory for read/modify/write operations
    float4 pos = mypos[atom];
    if (axis == 0) {
        tmppos[dof] = pos.x;
        pos.x += scaleFactor;
    } else if (axis == 1) {
        tmppos[dof] = pos.y;
        pos.y += scaleFactor;
    } else {
        tmppos[dof] = pos.z;
        pos.z += scaleFactor;
    }
    // Write back only once
    mypos[atom] = pos;
}