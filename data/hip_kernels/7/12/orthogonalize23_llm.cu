#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void orthogonalize23( float *Qi_gdof, int *blocksizes, int numblocks, int largestblock ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= numblocks) return; // Check bounds

    for (int j = 4; j < 6; j++) {
        for (int k = 3; k < j; k++) { // Vectors we're orthogonalizing against
            float dot_prod = 0.0;
            // Vectorized window into Qi_gdof
            float* base_j = Qi_gdof + i * 6 * largestblock + j;
            float* base_k = Qi_gdof + i * 6 * largestblock + k;

            // Use shared memory or loop unrolling for optimization
            for (int l = 0; l < blocksizes[i]; l++) {
                dot_prod += base_k[l * 6] * base_j[l * 6];
            }
            
            // Adjust Q_i
            for (int l = 0; l < blocksizes[i]; l++) {
                base_j[l * 6] -= base_k[l * 6] * dot_prod;
            }
        }

        float rotnorm = 0.0;
        // Norm calculation
        for (int l = 0; l < blocksizes[i]; l++) {
            float v = Qi_gdof[i * 6 * largestblock + l * 6 + j];
            rotnorm += v * v;
        }

        rotnorm = rsqrt(rotnorm); // Use rsqrt for faster computation

        // Normalize Q_i
        for (int l = 0; l < blocksizes[i]; l++) {
            Qi_gdof[i * 6 * largestblock + l * 6 + j] *= rotnorm;
        }
    }
}