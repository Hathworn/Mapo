#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void orthogonalize(float *eigvec, float *Qi_gdof, int cdof, int *blocksizes, int *blocknums, int largestblock) {
    int blockNum = blockIdx.x * blockDim.x + threadIdx.x;

    int curr_evec = 6;
    int size = blocksizes[blockNum];
    int startatom = blocknums[blockNum] / 3;

    // Cache current block offset in Qi_gdof for reuse
    int Qi_gdof_offset = blockNum * 6 * largestblock;

    for (int j = 0; j < size; j++) {
        if (curr_evec == size) break;

        for (int l = 0; l < size; l++) {
            Qi_gdof[Qi_gdof_offset + l * 6 + curr_evec] = eigvec[(blocknums[blockNum] + l) * largestblock + j];
        }

        for (int k = 0; k < curr_evec; k++) {
            double dot_prod = 0.0;
            for (int l = 0; l < size; l++) {
                dot_prod += Qi_gdof[Qi_gdof_offset + l * 6 + k] * eigvec[(blocknums[blockNum] + l) * largestblock + j];
            }

            // Fuse loops for updating Qi_gdof to decrease array accesses
            double *Qi_gdof_curr = Qi_gdof + Qi_gdof_offset + curr_evec;
            for (int l = 0; l < size; l++) {
                Qi_gdof_curr[l * 6] -= Qi_gdof[Qi_gdof_offset + l * 6 + k] * dot_prod;
            }
        }

        double norm = 0.0;
        for (int l = 0; l < size; l++) {
            norm += Qi_gdof[Qi_gdof_offset + l * 6 + curr_evec] * Qi_gdof[Qi_gdof_offset + l * 6 + curr_evec];
        }

        if (norm < 0.05) continue; // Skip if norm is too small

        norm = sqrt(norm);
        for (int l = 0; l < size; l++) {
            Qi_gdof[Qi_gdof_offset + l * 6 + curr_evec] /= norm;
        }

        curr_evec++;
    }

    for (int j = 0; j < curr_evec; j++) {
        for (int k = 0; k < size; k++) {
            eigvec[(startatom + k) * largestblock + (startatom + j)] = Qi_gdof[Qi_gdof_offset + k * 6 + j];
        }
    }
}