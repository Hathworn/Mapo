#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void makeBlockHessian(float *h, float *forces1, float *forces2, float *mass, float blockDelta, int *blocks, int *blocksizes, int numblocks, int *hessiannums, int *hessiansizes, int setnum, int N) {
    int blockNum = blockIdx.x * blockDim.x + threadIdx.x;
    if (blockNum >= numblocks) return; // Ensure within block range

    int dof = 3 * blocks[blockNum] + setnum;
    int atom = dof / 3;
    if (atom >= N || (blockNum != numblocks - 1 && atom >= blocks[blockNum + 1])) {
        return; // Out of bounds
    }

    int start_dof = 3 * blocks[blockNum];
    int end_dof = (blockNum == numblocks - 1) ? 3 * N : 3 * blocks[blockNum + 1];

    // Pre-compute block scale factor
    float invBlockDelta = 1.0 / blockDelta;

    for (int k = start_dof; k < end_dof; k++) {
        // Cache heavy calculations
        float massProduct = mass[atom] * mass[k / 3];
        float blockScale = invBlockDelta / sqrt(massProduct);

        // Compute index once
        int index = hessiannums[blockNum] + (k - start_dof) * (3 * blocksizes[blockNum]) + (dof - start_dof);

        // Optimize memory access pattern
        h[index] = (forces1[k] - forces2[k]) * blockScale;
    }
}