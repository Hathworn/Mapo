#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void counthits(int n, uint *hitsp, unsigned decalage_index) {
    // Optimize to reduce thread divergence and memory access latency
    hiprandStatePhilox4_32_10_t state;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned hits = 0;
    float x1, y1, x2, y2;

    // Initialize random state
    hiprand_init(0, index + decalage_index, 0, &state);

    for (int tries = 0; tries < n; tries += 2) {
        // Generate random numbers using hiprand
        float4 rand_vec = hiprand_uniform4(&state);
        x1 = 2 * rand_vec.x - 1;
        y1 = 2 * rand_vec.y - 1;
        x2 = 2 * rand_vec.z - 1;
        y2 = 2 * rand_vec.w - 1;

        // Accumulate hits for points inside unit circle
        hits += (x1 * x1 + y1 * y1 < 1) + (x2 * x2 + y2 * y2 < 1);
    }

    // Store hits count to global memory
    hitsp[index] = hits;
}