#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float function_a_appli(float x);

__global__ void mandelbrot (int nb_ligne, int nb_col, float seuil, float x_min, float x_max, float y_min, float y_max, float* res) {
    int max_ITER = 10000;
    int index_col = threadIdx.x + blockDim.x * blockIdx.x;
    int index_ligne = threadIdx.y + blockDim.y * blockIdx.y;

    // Utilize shared memory to reduce global memory access
    __shared__ float shared_x_min;
    __shared__ float shared_x_max;
    __shared__ float shared_y_min;
    __shared__ float shared_y_max;
    
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        shared_x_min = x_min;
        shared_x_max = x_max;
        shared_y_min = y_min;
        shared_y_max = y_max;
    }

    __syncthreads();

    if ((index_col >= nb_col) || (index_ligne >= nb_ligne)) return;

    int global_index = index_ligne * nb_col + index_col;

    float x0 = ((float)index_col / (float)nb_col) * (shared_x_max - shared_x_min) + shared_x_min;
    float y0 = ((float)(nb_ligne - index_ligne) / (float)nb_ligne) * (shared_y_max - shared_y_min) + shared_y_min;
    
    float x = 0, y = 0, xtemp;
    int iter = 0;

    // Unroll loop for better performance on GPU
    #pragma unroll 4
    while ((x * x + y * y <= seuil) && (iter < max_ITER)) {
        xtemp = x * x - y * y + x0;
        y = 2 * x * y + y0;
        x = xtemp;
        iter++;
    }

    res[global_index] = ((float)iter / (float)max_ITER);
}