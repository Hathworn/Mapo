#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prod(int taille, float* __restrict__ a, float b, float* __restrict__ c) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Ensure index remains within bounds
    if (index < taille) {
        c[index] = a[index] * b;
    }
}