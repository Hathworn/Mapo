#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void somme(int taille, float *a, float *b, float *c) {
    // Efficient index calculation
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early exit if index is out of bounds
    if (index < taille) {
        c[index] = a[index] + b[index];
    }
}