#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

long kMod = 1000000007;

__global__ void setData(int *dp, int n) {
    // Utilize shared memory and cooperative indexing for better performance
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    
    for(int i = idx; i < n; i += stride) {
        dp[i * n + i] = 1;
    }
}