#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

long kMod = 1000000007;

__global__ void prepareData(int *dp, int n) {
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x; // Calculate x index
    int idx_y = blockIdx.y * blockDim.y + threadIdx.y; // Calculate y index
    if (idx_x < n && idx_y < n) { // Check bounds
        dp[idx_x * n + idx_y] = 0; // Assign value
    }
}