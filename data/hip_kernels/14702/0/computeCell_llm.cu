#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void applyRule(char* left, char* middle, char* right, char* res) {
    char a = *left;
    char b = *middle;
    char c = *right;

    // Utilize bitwise operations for rule application
    unsigned char idx = (a << 2) | (b << 1) | c;
    unsigned char rule_output[8] = {0, 1, 1, 1, 0, 1, 1, 0};
    *res = rule_output[idx];
}

__global__ void computeCell(char* cellData, unsigned int* width, unsigned int* height) {
    int y = 0;
    int x = 0;

    for (y = 1; y < (*height); ++y) {
        for (x = threadIdx.x; x < (*width); x += blockDim.x) {
            char left = (x > 0) ? cellData[(x - 1) + (y - 1) * (*width)] : 0;
            char middle = cellData[x + (y - 1) * (*width)];
            char right = (x < (*width) - 1) ? cellData[(x + 1) + (y - 1) * (*width)] : 0;

            applyRule(&left, &middle, &right, &cellData[x + y * (*width)]);
        }
        __syncthreads();  // Synchronize threads at the end of each row
    }
}