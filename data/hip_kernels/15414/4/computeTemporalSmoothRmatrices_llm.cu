#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeTemporalSmoothRmatrices(const float* Rmatrices, uint32_t numSamples, uint32_t subArraySize, uint32_t numSubArrays, const uint32_t* subArraySizes, uint32_t temporalSmoothing, float* TempRmatrices)
{
    // Calculate thread index and sample index
    int tIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
    int sampleIdx = blockIdx.x;
    int scanlineIdxLocal = blockIdx.y;

    // Check if the current sample index is valid
    if (sampleIdx < numSamples)
    {
        // Get the local size of the sub-array
        int subArraySizeLocal = subArraySizes[scanlineIdxLocal * numSamples + sampleIdx];
        if (subArraySizeLocal > 0)
        {
            int numelR = subArraySizeLocal * (subArraySizeLocal + 1) / 2;
            int numelRfull = subArraySize * (subArraySize + 1) / 2;

            // Calculate the starting and ending indices for the temporal smoothing window
            int firstIdx = max(0, sampleIdx - (int)(temporalSmoothing)) + scanlineIdxLocal * numSamples;
            int lastIdx = min((int)(numSamples) - 1, sampleIdx + (int)(temporalSmoothing)) + scanlineIdxLocal * numSamples;

            float scaling = 1.0f;
            // Optimize by coalesced memory access and loop unrolling
            for (int matrixIdx = tIdx; matrixIdx < numelR; matrixIdx += blockDim.x * blockDim.y)
            {
                float finalEntry = 0.0f;
                for (int tempIdx = firstIdx; tempIdx <= lastIdx; tempIdx++)
                {
                    finalEntry += Rmatrices[matrixIdx + tempIdx * numelRfull];
                }
                // Write result to global memory
                TempRmatrices[matrixIdx + (scanlineIdxLocal * numSamples + sampleIdx) * numelRfull] = finalEntry * scaling;
            }
        }
    }
}