#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fillUnusedDiagonal(float* Rmatrices, uint32_t numSamples, uint32_t subArraySize, const uint32_t* subArraySizes)
{
    // Optimizing thread index calculation for readability
    int tIdx = threadIdx.y * blockDim.x + threadIdx.x;
    int sampleIdx = blockIdx.y * gridDim.x + blockIdx.x;

    // Avoid redundant calculations and memory reads
    if (sampleIdx < numSamples)
    {
        int subArraySizeLocal = subArraySizes[sampleIdx];
        if (subArraySize > subArraySizeLocal)
        {
            int numelRfull = subArraySize * subArraySize;
            float* R = &Rmatrices[sampleIdx * numelRfull];

            // Pre-fetch diagEntry outside the loop for efficiency
            float diagEntry = R[subArraySize * subArraySize - 1];

            // Optimize loop by avoiding calls to blockDim.x*blockDim.y within it
            const int stride = blockDim.x * blockDim.y;
            for (int diagIdx = subArraySizeLocal + tIdx; diagIdx < subArraySize; diagIdx += stride)
            {
                // Calculate index more efficiently
                int matrixIdx = diagIdx * (subArraySize + 1);

                R[matrixIdx] = diagEntry;
            }
        }
    }
}