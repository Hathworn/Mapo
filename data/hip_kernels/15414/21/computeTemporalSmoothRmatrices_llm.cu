#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeTemporalSmoothRmatrices(const float* Rmatrices, uint32_t numSamples, uint32_t subArraySize, uint32_t numSubArrays, const uint32_t* subArraySizes, uint32_t temporalSmoothing, float* TempRmatrices)
{
    // Calculate sample index based on grid and block index
    int sampleIdx = blockIdx.y * gridDim.x + blockIdx.x;

    // Check if the sample index is within the valid range
    if (sampleIdx < numSamples)
    {
        int subArraySizeLocal = subArraySizes[sampleIdx];
        int numelR = subArraySizeLocal * subArraySizeLocal;
        int numelRfull = subArraySize * subArraySize;

        // Determine the temporal smoothing range
        int firstIdx = max(0, sampleIdx - (int)temporalSmoothing);
        int lastIdx = min((int)(numSamples) - 1, sampleIdx + (int)temporalSmoothing);

        float scaling = 1.0f;

        // Optimize by unrolling shared processing across threads
        for (int matrixIdx = threadIdx.y * blockDim.x + threadIdx.x; matrixIdx < numelR; matrixIdx += blockDim.x * blockDim.y)
        {
            int colIdx = matrixIdx % subArraySizeLocal;
            int rowIdx = matrixIdx / subArraySizeLocal;
            int matrixStorageIdx = colIdx + rowIdx * subArraySize;

            float finalEntry = 0.0f;

            // Accumulate contributions from temporal smoothing range
            for (int tempIdx = firstIdx; tempIdx <= lastIdx; tempIdx++)
            {
                finalEntry += Rmatrices[matrixStorageIdx + tempIdx * numelRfull];
            }

            // Store the computed value in the temporary matrix output
            TempRmatrices[matrixStorageIdx + sampleIdx * numelRfull] = finalEntry * scaling;
        }
    }
}