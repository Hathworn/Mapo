#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    if (frame >= num_frames) return; // Boundary check for frame index

    float *feats = features + frame * ldf;
    
    // apply lifter coefficients using shared memory for lifter_coeffs
    __shared__ float shared_lifter_coeffs[CU1DBLOCK];
    if (thread_id < num_cols && cepstral_lifter != 0.0f) {
        shared_lifter_coeffs[thread_id] = lifter_coeffs[thread_id];
        __syncthreads(); // Ensure all threads have loaded lifter coefficients
        
        for (int c = thread_id; c < num_cols; c += CU1DBLOCK) {
            feats[c] *= shared_lifter_coeffs[c];
        }
    }
    
    // Thread 0 for each frame will apply energy
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = log(energy_floor);

        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        feats[0] = energy;
    }
}