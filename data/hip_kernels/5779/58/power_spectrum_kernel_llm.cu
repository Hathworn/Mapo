#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    int half_length = row_length / 2;

    if (thread_id < half_length) {
        if (thread_id == 0) {
            // Handle special case directly in kernel
            float real = A_in[blockIdx.x * ldi];
            float im = A_in[blockIdx.x * ldi + row_length];
            if (use_power) {
                A_out[blockIdx.x * ldo] = real * real;
                A_out[blockIdx.x * ldo + half_length] = im * im;
            } else {
                A_out[blockIdx.x * ldo] = fabs(real);
                A_out[blockIdx.x * ldo + half_length] = fabs(im);
            }
        } else {
            const float *Ar = A_in + blockIdx.x * ldi;
            float *Aw = A_out + blockIdx.x * ldo;
            float2 val = reinterpret_cast<const float2*>(Ar)[thread_id];
            float ret = val.x * val.x + val.y * val.y;
            Aw[thread_id] = use_power ? ret : sqrtf(ret);
        }
    }
}