#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    
    // Use shared memory for better performance
    __shared__ float shared_Ar[1024]; // adjust size based on requirement
    
    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        if (idx == 0) continue; // ignore special case
        
        shared_Ar[idx] = reinterpret_cast<float2 *>(Ar)[idx].x; // load data into shared memory
        shared_Ar[idx + half_length] = reinterpret_cast<float2 *>(Ar)[idx].y; // load data into shared memory
        
        float ret = shared_Ar[idx] * shared_Ar[idx] + shared_Ar[idx + half_length] * shared_Ar[idx + half_length];
        
        if (use_power) {
            Aw[idx] = ret; // store power
        } else {
            Aw[idx] = sqrtf(ret); // store magnitude
        }
    }

    // handle special case separately
    if (threadIdx.x == 0) {
        float real = shared_Ar[0] = Ar[0];  // load real part
        float im = shared_Ar[half_length] = Ar[row_length]; // load imaginary part
        
        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabs(real);
            Aw[half_length] = fabs(im);
        }
    }
}