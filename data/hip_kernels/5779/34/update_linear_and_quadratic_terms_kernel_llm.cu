#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    // Load current weight for consistent access
    float cur_weight = *cur_tot_weight;

    // Calculate new number of frames
    float new_num_frames = old_num_frames + cur_weight;
    float prior_scale_change = 1.0f;

    // Calculate prior scale change only if max_count is non-zero
    if (max_count != 0.0f) {
        float old_prior_scale = max(old_num_frames, (float)max_count) / max_count;
        float new_prior_scale = max(new_num_frames, (float)max_count) / max_count;
        prior_scale_change += new_prior_scale - old_prior_scale;
    }

    // Use threadIdx.x and blockIdx.x logic for iteration to avoid branching
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride = blockDim.x * gridDim.x;
    
    // Loop for processing each element, with i starting from the thread's unique index
    for (; i < n; i += stride) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        quadratic[diag_idx] += prior_scale_change;
    }

    // Update only once per grid, the condition ensures that only a single thread does this
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        linear[0] += prior_offset * prior_scale_change;
    }
}