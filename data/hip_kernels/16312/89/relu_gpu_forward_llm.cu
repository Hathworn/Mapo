#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Use grid-stride loop for improved memory access
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < N; i += stride) {
      // Minimize branching by using max function
      out[i] = max(in[i], 0.0f);
    }
}