#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if(id >= n * w * h) return; // Modified bounds check using total elements

    int b = id / (w * h * c); // Batch index
    int k = (id / (w * h)) % c; // Channel index
    int i = id % (w * h); // Spatial index

    int in_index = i + h*w*(k + b*c);
    int out_index = (k + c*b);

    // Perform reduction across spatial dimensions for avg pooling
    atomicAdd(&in_delta[in_index], out_delta[out_index] / (w*h)); 
}