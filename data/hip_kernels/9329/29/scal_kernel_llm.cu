#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index with 1D grid and block configuration
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by checking from i to i + blockDim.x * gridDim.x for continuous memory access
    for(; i < N; i += blockDim.x * gridDim.x)
    {
        X[i * INCX] *= ALPHA;
    }
}