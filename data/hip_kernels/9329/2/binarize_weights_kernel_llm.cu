#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;

    float mean = 0.0f;
    // Use shared memory to reduce global memory accesses
    extern __shared__ float shared_weights[];
    
    // Load weights into shared memory
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[i] = fabs(weights[f*size + i]);
    }
    __syncthreads();

    // Compute the mean using a single loop
    for (int i = 0; i < size; ++i) {
        mean += shared_weights[i];
    }
    mean /= size;

    // Binarize weights in parallel
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
    }
}