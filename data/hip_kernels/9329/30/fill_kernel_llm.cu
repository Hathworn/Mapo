#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Compute global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure each thread only processes its intended elements
    int stride = blockDim.x * gridDim.x;

    // Optimize loop for strided access
    for (; i < N; i += stride) {
        X[i * INCX] = ALPHA;
    }
}