#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    // Calculate global thread index
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Pre-compute indices for input and output
    int out_index = i;
    int out_w = out_index % (w*stride);
    int out_h = (out_index / (w*stride)) % (h*stride);
    int out_c = (out_index / (w*stride*h*stride)) % c;
    int b = out_index / (w*stride*h*stride*c);

    int in_w = out_w / stride;
    int in_h = out_h / stride;

    // Calculate input index
    int in_index = b*w*h*c + out_c*w*h + in_h*w + in_w;

    // Perform operation based on the direction flag (forward or backward)
    if (forward) {
        out[out_index] += scale * x[in_index];
    } else {
        atomicAdd(x + in_index, scale * out[out_index]);
    }
}