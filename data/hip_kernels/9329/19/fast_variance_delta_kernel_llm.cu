#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta) 
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Use shared memory to store precalculated values to reduce redundant calculation
    __shared__ float mean_shared, variance_shared;

    if (id == 0) {
        mean_shared = mean[filter];
        variance_shared = variance[filter];
    }
    __syncthreads();

    int i, j;
    for(j = 0; j < batch; ++j) {
        for(i = 0; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i + id;

            // Load values to avoid redundant access and perform calculation
            float x_val = (i + id < spatial) ? x[index] : 0;
            float delta_val = (i + id < spatial) ? delta[index] : 0;
            
            local[id] += delta_val * (x_val - mean_shared);
        }
    }
    __syncthreads();

    // Use atomic operation for reduction to enhance performance
    atomicAdd(&variance_delta[filter], local[id]);

    if(id == 0) {
        variance_delta[filter] *= -.5 * powf(variance_shared + .000001f, (float)(-3./2.));
    }
}