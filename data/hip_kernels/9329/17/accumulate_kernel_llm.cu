#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Calculate global thread index
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Initialize sum to zero using local variable
    float temp_sum = 0.0f;

    // Iterate over elements with stride based on groups
    for(int k = 0; k < n; ++k) {
        temp_sum += x[k * groups + i];
    }

    // Write result back to global memory
    sum[i] = temp_sum;
}