#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1.F / (batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float sum = 0.0f;  // Use a temporary variable to accumulate
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            float diff = x[index] - mean[i];
            sum += diff * diff;  // Reduce redundant 'powf' call
        }
    }
    variance[i] = sum * scale;  // Scale after accumulation
}