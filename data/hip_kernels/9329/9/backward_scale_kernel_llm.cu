#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Use a single loop over all elements
    for(int b = 0; b < batch; ++b){
        for(int i = threadIdx.x; i < size; i += BLOCK){
            int index = i + size*(filter + n*b);
            sum += delta[index] * x_norm[index];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Use atomicAdd to avoid race condition 
    if (p == 0) {
        float blockSum = 0;
        for(int i = 0; i < BLOCK; ++i) {
            blockSum += part[i];
        }
        atomicAdd(&scale_updates[filter], blockSum);
    }
}