#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Use shared memory to minimize global memory access
    extern __shared__ float shared_biases[];
    if (threadIdx.x == 0) {
        shared_biases[filter] = biases[filter];
    }
    __syncthreads();

    if(offset < size) {
        output[(batch*n+filter)*size + offset] += shared_biases[filter];
    }
}