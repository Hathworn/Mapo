#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplify ID calculation (assuming 1D grid)
    if(id < size) {
        float rand_val = rand[id]; // Cache read value from global memory
        input[id] = (rand_val < prob) ? 0.0f : input[id] * scale; // Simplify ternary operation
    }
}