#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Calculate the global index, accounting for grid and block dimensions
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Stride over elements to allow handling of more elements per block
    int stride = gridDim.x * blockDim.x;
    
    for (; i < N; i += stride) {
        // Perform power operation for the given index and store result
        Y[i * INCY] = powf(X[i * INCX], ALPHA);
    }
}