#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate unique global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Perform calculations only if index is within bounds
    if (i < n) {
        float dci = dc[i]; // Pre-load dc[i] to reduce redundant memory access
        float si = s[i];   // Pre-load s[i] to reduce redundant memory access
        
        if (da) da[i] += dci * si;
        db[i] += dci * (1.0f - si);
        ds[i] += dci * (a[i] - b[i]);
    }
}