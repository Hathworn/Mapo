#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    // Calculate thread index
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    
    // Return if out of bounds
    if (i >= filters) return;

    // Initialize mean_delta to 0 for each filter
    float sum = 0.0f;
    for (int j = 0; j < batch * spatial; ++j) {
        // Compute linear index and accumulate delta values
        int index = j * filters + i;
        sum += delta[index];
    }

    // Normalize and store result in mean_delta
    mean_delta[i] = sum * (-1.0f / sqrtf(variance[i] + 0.000001f));
}