#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Flatten blockIdx
    int stride = gridDim.x * blockDim.x;           // Compute stride for loop

    for (; i < N; i += stride) {                   // Loop over elements with stride
        X[i * INCX] = ALPHA;                        // Assign ALPHA
    }
}