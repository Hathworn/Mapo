#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0;

    // Faster loop using warp divergence elimination
    for (int j = 0; j < batch; ++j) {
        int baseIdx = j * spatial * filters + filter * spatial;
        for (int i = id; i < spatial; i += threads) {
            int index = baseIdx + i;
            local[id] += delta[index];
        }
    }
    __syncthreads();

    // Reduce sum using shared memory with fewer iterations
    if (id < warpSize) {
        for (int i = id + warpSize; i < threads; i += warpSize) {
            local[id] += local[i];
        }
    }
    __syncthreads();

    // Final computation by the first thread
    if (id == 0) {
        mean_delta[filter] = local[0] * (-1.F / sqrtf(variance[filter] + .000001f));
    }
}