#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Optimize by using a single loop with fewer conditional checks
    int stride = blockDim.x * gridDim.x;
    for (int j = id; j < batch * spatial; j += stride) {
        int index = j * filters + filter * spatial + id;
        if (index < batch * spatial * filters) {
            local[id] += x[index];
        }
    }
    __syncthreads();

    // Use warp reduction for faster summation
    if (threads >= 32) {
        for (int s = threads / 2; s > 0; s >>= 1) {
            if (id < s) {
                local[id] += local[id + s];
            }
            __syncthreads();
        }
    }

    // Single thread division after warp reduction
    if (id == 0) {
        mean[filter] = local[0] / (spatial * batch);
    }
}