#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    float sum = 0;
    float largest = -INFINITY;

    // Use a single loop to find the largest element and compute exponentials
    for(int i = 0; i < n; ++i){
        float val = input[i];
        largest = (val > largest) ? val : largest;
    }
    float exp_values[n];
    for(int i = 0; i < n; ++i){
        exp_values[i] = exp((input[i] - largest) / temp);
        sum += exp_values[i];
    }
 
    // Normalize the output in the same loop for better cache locality
    for(int i = 0; i < n; ++i){
        output[i] = exp_values[i] / sum;
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if(b < batch) {
        softmax_device(n, input + b * offset, temp, output + b * offset);
    }
}