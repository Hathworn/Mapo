#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    // Use stride loop for better utilization of GPU resources
    while(i < N) {
        X[i*INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i*INCX]));
        i += blockDim.x * gridDim.x; // Move to the next index this thread should handle
    }
}