#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation
    if (idx < N) {
        Y[idx * INCY + OFFY] = X[idx * INCX + OFFX]; // Access elements directly
    }
}