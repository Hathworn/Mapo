#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    // Define shared memory per block to store temporary results for reduction
    extern __shared__ float shared_deltas[];

    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;
    int area = (size - 1) / stride;

    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int index = id;
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0;
    int l, m;

    // Load shared memory with initial zeroed values
    for (int i = threadIdx.x; i < blockDim.x; i += blockDim.x) {
        shared_deltas[i] = 0.0f;
    }
    __syncthreads();

    // Accumulate deltas into shared memory
    for (l = -area; l < area + 1; ++l) {
        for (m = -area; m < area + 1; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            int out_index = out_w + w * (out_h + h * (k + c * b));
            int valid = (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h);
            if (valid && indexes[out_index] == index) {
                atomicAdd(&shared_deltas[threadIdx.x], delta[out_index]);
            }
        }
    }

    __syncthreads();

    // Reduce the shared memory values back into global memory
    if (threadIdx.x == 0) {
        float block_d = 0.0f;
        for (int i = 0; i < blockDim.x; i++) {
            block_d += shared_deltas[i];
        }
        atomicAdd(&prev_delta[index], block_d);
    }
}