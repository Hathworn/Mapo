#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + 2*pad)/stride;
    int w = (in_w + 2*pad)/stride;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    // calculate initial indices
    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % in_c;
    int b = id / (w * h * in_c);

    int out_index = j + w*(i + h*(k + in_c*b));
    float max = -INFINITY;
    int max_i = -1;

    int w_offset = -pad;
    int h_offset = -pad;

    // combine the loops for better cache locality
    for(int l = 0; l < size; ++l) {
        int cur_h = h_offset + i*stride + l;
        for(int m = 0; m < size; ++m) {
            int cur_w = w_offset + j*stride + m;
            int index = cur_w + in_w*(cur_h + in_h*(k + b*in_c));
            if (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w) {
                float val = input[index];
                if(val > max) { // directly update max and max_i
                    max = val;
                    max_i = index;
                }
            }
        }
    }
    output[out_index] = max;
    indexes[out_index] = max_i;
}