#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check index within bounds
    if (i < n) {
        // Perform binarization
        binary[i] = (x[i] >= 0) ? 1 : -1;
    }
}