#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;

    // Precompute offset for index calculation
    int size_n = size * n;

    for(int b = 0; b < batch; ++b) {
        for(int i = 0; i < size; ++i) {
            int base = b * size_n;
            int i1 = base + f1 * size + i;
            int i2 = base + f2 * size + i;
            float output1 = output[i1];
            float output2 = output[i2];
            sum += output1 * output2;
            norm1 += output1 * output1;
            norm2 += output2 * output2;
        }
    }

    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    float scaled_sum = -scale * sum / norm;

    for(int b = 0; b < batch; ++b) {
        for(int i = 0; i < size; ++i) {
            int base = b * size_n;
            int i1 = base + f1 * size + i;
            int i2 = base + f2 * size + i;
            delta[i1] += scaled_sum * output[i2];
            delta[i2] += scaled_sum * output[i1];
        }
    }
}