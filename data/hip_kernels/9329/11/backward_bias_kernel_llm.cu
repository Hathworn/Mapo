#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Loop unrolling for improved performance
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; i += BLOCK * 4) {
            int index1 = p + i + size * (filter + n * b);
            int index2 = index1 + BLOCK;
            int index3 = index1 + 2 * BLOCK;
            int index4 = index1 + 3 * BLOCK;
            if (p + i < size) sum += delta[index1];
            if (p + i + BLOCK < size) sum += delta[index2];
            if (p + i + 2 * BLOCK < size) sum += delta[index3];
            if (p + i + 3 * BLOCK < size) sum += delta[index4];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Use a parallel reduction to accumulate the results
    if (BLOCK >= 512) { if (p < 256) part[p] += part[p + 256]; __syncthreads(); }
    if (BLOCK >= 256) { if (p < 128) part[p] += part[p + 128]; __syncthreads(); }
    if (BLOCK >= 128) { if (p < 64) part[p] += part[p + 64]; __syncthreads(); }

    if (p < 32) {
        // Warp reduction without syncthreads
        volatile float *vpart = part;
        if (BLOCK >= 64) vpart[p] += vpart[p + 32];
        if (BLOCK >= 32) vpart[p] += vpart[p + 16];
        if (BLOCK >= 16) vpart[p] += vpart[p + 8];
        if (BLOCK >= 8) vpart[p] += vpart[p + 4];
        if (BLOCK >= 4) vpart[p] += vpart[p + 2];
        if (BLOCK >= 2) vpart[p] += vpart[p + 1];
    }

    if (p == 0) bias_updates[filter] += part[0];
}