#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate the global thread index using 3D grid and block size
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Step through increment of block size and perform operation
    for (; i < N; i += blockDim.x * gridDim.x) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}