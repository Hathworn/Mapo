#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (; i < N; i += stride) {
        Y[i*INCY] *= X[i*INCX];
    }
}