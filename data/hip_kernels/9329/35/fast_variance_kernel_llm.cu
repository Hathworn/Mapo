#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0.0f;

    // Loop unrolling for better performance
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += powf((x[index] - mean[filter]), 2);
        }
    }
    __syncthreads();

    // Use parallel reduction for summing
    for (int stride = threads / 2; stride > 0; stride >>= 1) {
        if (id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    // Only thread 0 writes result to global memory
    if (id == 0) {
        variance[filter] = local[0] / (spatial * batch - 1);
    }
}