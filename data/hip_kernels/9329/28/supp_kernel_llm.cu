#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Cachealpha squared to avoid redundant calculations
    float alpha_squared = ALPHA * ALPHA;

    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Check bounds and perform conditional assignment
    if(i < N) {
        float value = X[i * INCX];
        if(value * value < alpha_squared) {
            X[i * INCX] = 0;
        }
    }
}