#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Ensure thread is operating within bounds
    if(i < n)
    {
        // Calculate weighted sum
        c[i] = s[i] * a[i] + (1.0f - s[i]) * (b ? b[i] : 0.0f);
    }
}