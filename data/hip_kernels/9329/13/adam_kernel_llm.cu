#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    // Use shared memory to optimize repetitive calculations
    __shared__ float shared_B1_pow, shared_B2_pow;
    if (threadIdx.x == 0) {
        shared_B1_pow = 1.F - powf(B1, t);
        shared_B2_pow = sqrtf(1.F - powf(B2, t));
    }
    __syncthreads();

    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index < N) {
        // Reuse shared memory values for computation
        float denom = shared_B1_pow * (sqrtf(v[index]) + eps);
        x[index] -= rate * shared_B2_pow * m[index] / denom;
    }
}