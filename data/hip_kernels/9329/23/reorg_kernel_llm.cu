#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    // Calculate parameters for input and output indices
    int b = index / (w * h * c);
    int in_index = index % (w * h * c);
    int in_c = in_index / (w * h);
    in_index %= (w * h);
    int in_h = in_index / w;
    int in_w = in_index % w;

    int out_c = c / (stride * stride);
    int offset = in_c / out_c;
    int c2 = in_c % out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    int out_index = ((b * out_c + c2) * h * stride + h2) * w * stride + w2;

    // Simplified conditional data transfer
    if (forward) {
        out[out_index] = x[index];
    } else {
        out[index] = x[out_index];
    }
}