#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilize warp divergence minimization
    if (i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff); // Use fast math function for absolute value

        if (abs_val < 1.0f) {
            error[i] = diff * diff;
            delta[i] = diff;
        } else {
            error[i] = 2.0f * abs_val - 1.0f;
            delta[i] = copysignf(1.0f, diff); // Use built-in function for sign
        }
    }
}