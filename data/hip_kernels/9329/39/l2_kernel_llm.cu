#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use only blockIdx.x for 1D grid
    if(i < n) { 
        float diff = truth[i] - pred[i];
        error[i] = diff * diff;
        delta[i] = diff;
    }
}