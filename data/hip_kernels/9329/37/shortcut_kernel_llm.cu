#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    // Calculate unique thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return;
    
    // Leverage modulo and division for multi-dimensional indexing
    int temp_id = id;
    int i = temp_id % minw;
    temp_id /= minw;
    int j = temp_id % minh;
    temp_id /= minh;
    int k = temp_id % minc;
    temp_id /= minc;
    int b = temp_id;  // Direct assignment for batch index
    
    // Precompute strides for efficient indexing
    int out_index = i*sample + w2*(j*sample + h2*(k + c2*b));
    int add_index = i*stride + w1*(j*stride + h1*(k + c1*b));
    
    // Optimized addition operation
    out[out_index] += add[add_index];
}