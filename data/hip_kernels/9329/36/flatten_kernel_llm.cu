#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N) return;
    
    // Calculate indices using modulo and integer division
    int in_s = i % spatial;
    int in_c = (i / spatial) % layers;
    int b = i / (spatial * layers);

    // Calculate flattened indices for input and output
    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;

    // Assign elements based on the forward flag
    if (forward) out[i2] = x[i1];
    else out[i1] = x[i2];
}