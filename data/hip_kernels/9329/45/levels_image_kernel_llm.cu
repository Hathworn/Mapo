#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float3 hsv_to_rgb_kernel(float3 hsv) {
    float h = hsv.x, s = hsv.y, v = hsv.z;
    float r, g, b, f, p, q, t;
    int index = static_cast<int>(floorf(h));
    
    f = h - index;
    p = v * (1 - s);
    q = v * (1 - s * f);
    t = v * (1 - s * (1 - f));

    // Use a switch statement for clearer and potentially more efficient branching
    switch (index) {
        case 0: r = v; g = t; b = p; break;
        case 1: r = q; g = v; b = p; break;
        case 2: r = p; g = v; b = t; break;
        case 3: r = p; g = q; b = v; break;
        case 4: r = t; g = p; b = v; break;
        default: r = v; g = p; b = q; break;
    }

    // Clamp the values to [0,1]
    r = fminf(fmaxf(r, 0.0f), 1.0f);
    g = fminf(fmaxf(g, 0.0f), 1.0f);
    b = fminf(fmaxf(b, 0.0f), 1.0f);

    return make_float3(r, g, b);
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb) {
    float r = rgb.x, g = rgb.y, b = rgb.z;
    float h, s, v, max, min, delta;

    max = fmaxf(fmaxf(r, g), b);
    min = fminf(fminf(r, g), b);
    delta = max - min;
    v = max;

    if (max != 0) {
        s = delta / max;
        if (r == max) {
            h = (g - b) / delta;
        } else if (g == max) {
            h = 2 + (b - r) / delta;
        } else {
            h = 4 + (r - g) / delta;
        }
        if (h < 0) h += 6;
    } else {
        s = 0;
        h = -1;
    }

    return make_float3(h, s, v);
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift) {
    int size = batch * w * h;
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    if (id >= size) return;
    
    int x = id % w;
    int y = (id / w) % h;
    int b = id / (w * h);

    // Pre-calculate values that are reused
    float rshift = rand[0], gshift = rand[1], bshift = rand[2];
    float r0 = rand[8 * b + 0], r1 = rand[8 * b + 1];
    float r2 = rand[8 * b + 2], r3 = rand[8 * b + 3];

    saturation = (r0 * (saturation - 1) + 1);
    if (r1 > 0.5f) saturation = 1.0f / saturation;
    
    exposure = (r2 * (exposure - 1) + 1);
    if (r3 > 0.5f) exposure = 1.0f / exposure;

    size_t offset = (b * h + y) * w + x;
    float r = image[offset];
    float g = image[offset + w];
    float b = image[offset + 2 * w];
    float3 rgb = make_float3(r, g, b);

    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0.0f;
    }
    
    image[offset] = fmaf(rgb.x, scale, translate + (rshift - .5f) * shift);
    image[offset + w] = fmaf(rgb.y, scale, translate + (gshift - .5f) * shift);
    image[offset + 2 * w] = fmaf(rgb.z, scale, translate + (bshift - .5f) * shift);
}