#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    // Calculate the unique global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within bounds
    if (index < N) {
        // Compute filter index using modulus operation directly
        int f = (index / spatial) % filters;
        
        // Normalize the element using precomputed mean and variance
        x[index] = (x[index] - mean[f]) / (sqrtf(variance[f]) + 1e-6f);
    }
}