#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mask_kernel(int n,  float *x, float mask_num, float *mask)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use grid-stride loop for better utilization
    for (; i < n; i += blockDim.x * gridDim.x)
    {
        if (mask[i] == mask_num)
        {
            x[i] = mask_num;
        }
    }
}