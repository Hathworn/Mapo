#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.F / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use block-level reduction
    __shared__ float shared_mean[1024]; // Adjust size according to max threads per block
    shared_mean[threadIdx.x] = 0.0f;
    
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            shared_mean[threadIdx.x] += x[index]; // Aggregate in shared memory
        }
    }
    
    // Synchronize threads to ensure all additions are complete
    __syncthreads();
    
    // Reduce results within a block
    if (threadIdx.x == 0) {
        float block_total = 0.0f;
        for (int n = 0; n < blockDim.x; ++n) {
            block_total += shared_mean[n];
        }
        atomicAdd(&mean[i / blockDim.x], block_total * scale); // Atomic add to global result
    }
}