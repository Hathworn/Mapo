#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Compute the thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Initialize shared memory for variance_delta
    __shared__ float shared_variance_delta[1024];  // Assuming a block size limit for optimization
    shared_variance_delta[threadIdx.x] = 0;

    // Main loop over batch and spatial dimensions
    for (int j = 0; j < batch; ++j)
    {
        for (int k = 0; k < spatial; ++k)
        {
            int index = j * filters * spatial + i * spatial + k;
            shared_variance_delta[threadIdx.x] += delta[index] * (x[index] - mean[i]);
        }
    }

    // Store the weighted variance delta in shared memory
    shared_variance_delta[threadIdx.x] *= -.5f * powf(variance[i] + .000001f, -1.5f);

    // Write final result from shared memory back to global memory
    variance_delta[i] = shared_variance_delta[threadIdx.x];
}