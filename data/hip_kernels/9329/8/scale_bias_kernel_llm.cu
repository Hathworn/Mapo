#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    // Calculate global position based on three-dimensional block and thread indices
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Check if offset is within range, then process
    if(offset < size) {
        int index = (batch * n + filter) * size + offset; // Combine calculations into single step
        output[index] *= biases[filter];
    }
}