#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = blockIdx.x * blockDim.x + threadIdx.x;
    if (s >= size) return;

    // Optimize the mean calculation by reducing repeated index computations
    float mean = 0;
    int idx_base = s;
    for(int i = 0; i < n; ++i){
        mean += fabs(input[idx_base]);
        idx_base += size;
    }
    mean /= n;

    // Recompute base index and binarize
    idx_base = s;
    for(int i = 0; i < n; ++i){
        binary[idx_base] = (input[idx_base] > 0) ? mean : -mean;
        idx_base += size;
    }
}