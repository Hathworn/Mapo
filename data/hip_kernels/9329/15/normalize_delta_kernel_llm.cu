#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        int f = (index / spatial) % filters;

        // Precompute common expressions to reduce redundant calculations
        float variance_sqrt_inv = 1.F / (sqrtf(variance[f]) + 0.000001f);
        float spatial_batch_factor = 1.0f / (spatial * batch);
        float x_mean_diff = x[index] - mean[f];

        delta[index] = delta[index] * variance_sqrt_inv + 
                       variance_delta[f] * 2.0f * x_mean_diff * spatial_batch_factor + 
                       mean_delta[f] * spatial_batch_factor;
    }
}