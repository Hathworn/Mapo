#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate global index using 1D grid and block layout
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread accesses only valid array indices
    if(i < n){
        c[i] += a[i] * b[i];
    }
}