#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bgr_to_gray_kernel(unsigned char* input, unsigned char* output, int width, int height, int colorWidthStep, int grayWidthStep) {
    // Calculate the unique thread ID
    const int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    // Perform computation only if within bounds
    if (xIndex < width && yIndex < height) {
        // Compute single index offset for input and output
        const int color_tid = yIndex * colorWidthStep + (3 * xIndex);
        const int gray_tid = yIndex * grayWidthStep + xIndex;

        // Load BGR values from global memory
        const unsigned char blue = input[color_tid];
        const unsigned char green = input[color_tid + 1];
        const unsigned char red = input[color_tid + 2];

        // Convert BGR to grayscale using NTSC formula
        const float gray = red * 0.3f + green * 0.59f + blue * 0.11f;

        // Store result in output as a grayscale pixel
        output[gray_tid] = static_cast<unsigned char>(gray);
    }
}