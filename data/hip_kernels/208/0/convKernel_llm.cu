#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convKernel(const float* __restrict__ source, const float* __restrict__ kernel, float* __restrict__ target, const int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= len) return;

    float value = 0.0f;

    // Unrolling the loop for better performance
    #pragma unroll 4
    for (int i = 0; i < len; i++) {
        value += source[i] * kernel[(len + len / 2 + idx - i) % len]; // Positive modulo
    }

    target[idx] = value;
}