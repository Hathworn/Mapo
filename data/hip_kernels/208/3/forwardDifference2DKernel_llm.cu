#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forwardDifference2DKernel(const int cols, const int rows, const float* data, float* dx, float* dy) {
    // Use shared memory to enhance memory access patterns
    extern __shared__ float shared_data[];

    // Calculate global and shared memory indices
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int global_idy = blockIdx.y * blockDim.y + threadIdx.y;
    int local_idx = threadIdx.x;
    int local_idy = threadIdx.y;

    if (global_idx < cols && global_idy < rows) {
        // Load data into shared memory
        shared_data[local_idy * blockDim.x + local_idx] = data[global_idy * cols + global_idx];
    }
    __syncthreads(); // Ensure all data are loaded into shared memory

    if (global_idx < cols - 1 && global_idy < rows - 1) {
        // Calculate forward difference if within bounds
        if (local_idx < blockDim.x - 1) {
            dx[global_idy * cols + global_idx] = shared_data[local_idy * blockDim.x + local_idx + 1] - shared_data[local_idy * blockDim.x + local_idx];
        } else {
            dx[global_idy * cols + global_idx] = data[global_idy * cols + global_idx + 1] - data[global_idy * cols + global_idx];
        }

        if (local_idy < blockDim.y - 1) {
            dy[global_idy * cols + global_idx] = shared_data[(local_idy + 1) * blockDim.x + local_idx] - shared_data[local_idy * blockDim.x + local_idx];
        } else {
            dy[global_idy * cols + global_idx] = data[(global_idy + 1) * cols + global_idx] - data[global_idy * cols + global_idx];
        }
    }
}