#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forwardDifference2DAdjointKernel(const int cols, const int rows, const float* dx, const float* dy, float* target) {
    // Use shared memory for better performance; allocate space for a tile
    __shared__ float s_dx[TILE_DIM + 1][TILE_DIM + 1];
    __shared__ float s_dy[TILE_DIM + 1][TILE_DIM + 1];

    // Calculating global indices
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int global_idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Load data into shared memory with boundary checks
    if (global_idx < rows && global_idy < cols) {
        s_dx[threadIdx.y][threadIdx.x] = dx[global_idy * rows + global_idx];
        s_dy[threadIdx.y][threadIdx.x] = dy[global_idy * rows + global_idx];
    }
    __syncthreads();

    // Processing central elements of the block (avoids boundary overwrite)
    if (global_idx > 0 && global_idx < rows - 1 && global_idy > 0 && global_idy < cols - 1) {
        const auto index = global_idy * rows + global_idx;
        target[index] = -s_dx[threadIdx.y][threadIdx.x] + s_dx[threadIdx.y][threadIdx.x - 1]
                        - s_dy[threadIdx.y][threadIdx.x] + s_dy[threadIdx.y - 1][threadIdx.x];
    }
}