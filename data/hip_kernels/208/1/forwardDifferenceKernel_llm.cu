#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forwardDifferenceKernel(const int len, const float* __restrict__ source, float* __restrict__ target) {
    // Use shared memory for faster access
    extern __shared__ float shared_source[];

    int localIdx = threadIdx.x + 1;
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if (globalIdx < len - 1) {
        // Load element into shared memory
        shared_source[localIdx] = source[globalIdx - 1];
        shared_source[localIdx + 1] = source[globalIdx];

        __syncthreads();

        // Compute forward difference using shared memory
        target[globalIdx] = shared_source[localIdx + 1] - shared_source[localIdx];
    }
}