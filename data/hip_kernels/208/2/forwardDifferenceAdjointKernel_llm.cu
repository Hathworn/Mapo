#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forwardDifferenceAdjointKernel(const int len, const float* source, float* target) {
    // Use shared memory for better memory access patterns
    __shared__ float sharedSource[1024];
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;

    if (idx < len - 1) {
        sharedSource[threadIdx.x] = source[idx - 1];
        __syncthreads();

        // Avoids extra out-of-bound checks and utilizes shared memory
        target[idx] = -source[idx] + sharedSource[threadIdx.x];
    }
}