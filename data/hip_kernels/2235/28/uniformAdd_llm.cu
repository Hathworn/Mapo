#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    // Load uniform value to shared memory for each block
    __shared__ float uni;
    if (threadIdx.x == 0) {
        uni = uniforms[blockIdx.x + blockOffset];
    }

    // Calculate global index
    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x;

    // Ensure previous stores to shared memory are visible to threads
    __syncthreads();
    
    // Perform addition if within bounds
    if (address < n) {
        g_data[address] += uni;
    }
    
    // Execute second addition only for valid threads
    unsigned int secondAddress = address + blockDim.x;
    if (secondAddress < n) {
        g_data[secondAddress] += uni;
    }
}