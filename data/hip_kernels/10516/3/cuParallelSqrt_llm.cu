#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Constants used by the program
#define BLOCK_DIM                      16

//-----------------------------------------------------------------------------------------------//
//                                            KERNELS                                            //
//-----------------------------------------------------------------------------------------------//

/**
* Computes the square root of the first line (width-th first element)
* of the distance matrix.
*
* @param dist    distance matrix
* @param width   width of the distance matrix
* @param k       number of neighbors to consider
*/
__global__ void cuParallelSqrt(float *dist, int width, int k) {
    // Use 1D block and grid to improve memory coalescing
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (xIndex < width * k) {
        dist[xIndex] = sqrt(dist[xIndex]);
    }
}