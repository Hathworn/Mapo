#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Constants used by the program
#define BLOCK_DIM                      16

// Kernel function for computing distances
__global__ void cuComputeDistanceGlobal(float* A, int wA, float* B, int wB, int dim, float* AB) {
    // Shared memory for sub-matrices of A and B
    __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Initialize variables
    float ssd = 0;

    // Calculate start positions
    int begin_A = BLOCK_DIM * blockIdx.y * wA;
    int begin_B = BLOCK_DIM * blockIdx.x * wB;
    int end_A = begin_A + (dim - 1) * wA;

    // Conditions
    bool cond0 = (begin_A + tx < wA * dim);  // Condition for matrix A
    bool cond1 = (begin_B + tx < wB * dim);  // Condition for matrix B

    // Iterate over sub-matrices
    for (int a = begin_A, b = begin_B; a <= end_A; a += BLOCK_DIM * wA, b += BLOCK_DIM * wB) {
        // Load data to shared memory
        shared_A[ty][tx] = (cond0 && (a / wA + ty) < dim) ? A[a + wA * ty + tx] : 0;
        shared_B[ty][tx] = (cond1 && (b / wB + ty) < dim) ? B[b + wB * ty + tx] : 0;

        // Synchronize to ensure shared memory loading is complete
        __syncthreads();

        // Compute squared differences and accumulate
        for (int k = 0; k < BLOCK_DIM; ++k) {
            float diff = shared_A[k][ty] - shared_B[k][tx];
            ssd += diff * diff;
        }

        // Synchronize before loading next sub-matrices
        __syncthreads();
    }

    // Write result to global memory
    if (cond0 && cond1) {
        AB[(BLOCK_DIM * blockIdx.y + ty) * wB + BLOCK_DIM * blockIdx.x + tx] = ssd;
    }
}