#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK_DIM 16

__global__ void cuInsertionSort(float *dist, int *ind, int width, int height, int k) {
    // Get unique thread index
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (xIndex < width) {
        float *p_dist = dist + xIndex;
        int *p_ind = ind + xIndex;
        float max_dist = p_dist[0];
        p_ind[0] = 1;

        // Part 1: Sort first k elements
        for (int l = 1; l < k; l++) {
            int curr_row = l * width;
            float curr_dist = p_dist[curr_row];
            if (curr_dist < max_dist) {
                int i = l - 1;
                for (int a = 0; a < l - 1; a++) {
                    if (p_dist[a * width] > curr_dist) {
                        i = a;
                        break;
                    }
                }
                for (int j = l; j > i; j--) {
                    p_dist[j * width] = p_dist[(j - 1) * width];
                    p_ind[j * width] = p_ind[(j - 1) * width];
                }
                p_dist[i * width] = curr_dist;
                p_ind[i * width] = l + 1;
            } else {
                p_ind[l * width] = l + 1;
            }
            // Update max distance
            max_dist = p_dist[curr_row];
        }

        // Part 2: Insert elements into k-th sorted elements
        int max_row = (k - 1) * width;
        for (int l = k; l < height; l++) {
            float curr_dist = p_dist[l * width];
            if (curr_dist < max_dist) {
                int i = k - 1;
                for (int a = 0; a < k - 1; a++) {
                    if (p_dist[a * width] > curr_dist) {
                        i = a;
                        break;
                    }
                }
                for (int j = k - 1; j > i; j--) {
                    p_dist[j * width] = p_dist[(j - 1) * width];
                    p_ind[j * width] = p_ind[(j - 1) * width];
                }
                p_dist[i * width] = curr_dist;
                p_ind[i * width] = l + 1;
                // Update max distance
                max_dist = p_dist[max_row];
            }
        }
    }
}