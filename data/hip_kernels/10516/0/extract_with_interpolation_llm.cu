#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_DIM 16

__global__ void extract_with_interpolation(int nthreads, float *data, float *n_xy_coords, float *extracted_data, int n_max_coord, int channels, int height, int width) {
    // Calculate the unique thread index within the grid
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < nthreads) {
        int n = index / n_max_coord;
        int nd = n * n_max_coord * channels;
        float x = n_xy_coords[index * 2];
        float y = n_xy_coords[index * 2 + 1];

        // Calculate floor values and bounds check
        int x0 = max(0, min(width - 1, static_cast<int>(floor(x))));
        int y0 = max(0, min(height - 1, static_cast<int>(floor(y))));
        int x1 = min(width - 1, x0 + 1);
        int y1 = min(height - 1, y0 + 1);

        // Calculate weights for interpolation
        float wx0 = static_cast<float>(x1) - x;
        float wx1 = x - x0;
        float wy0 = static_cast<float>(y1) - y;
        float wy1 = y - y0;

        if (x0 == x1) { wx0 = 1; wx1 = 0; }
        if (y0 == y1) { wy0 = 1; wy1 = 0; }

        for (int c = 0; c < channels; c++) {
            int nc = (n * channels + c) * height;
            // Bilinear interpolation formula
            extracted_data[nd + index % n_max_coord + n_max_coord * c] = 
                wy0 * wx0 * data[(nc + y0) * width + x0] +
                wy1 * wx0 * data[(nc + y1) * width + x0] +
                wy0 * wx1 * data[(nc + y0) * width + x1] +
                wy1 * wx1 * data[(nc + y1) * width + x1];
        }
    }
}