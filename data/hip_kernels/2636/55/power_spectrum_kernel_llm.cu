#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    const float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    int stride = blockDim.x;

    // Optimization: Use a stride loop to allow more threads to participate
    for (int idx = thread_id; idx < half_length; idx += stride) {
        if (idx == 0) continue;

        float2 val = reinterpret_cast<const float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }

    // Synchronize threads before handling the special case
    __syncthreads();

    // Handle special case with single thread
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];
        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}