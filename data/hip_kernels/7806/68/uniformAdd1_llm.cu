#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd1(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex)
{
    __shared__ int uni;
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];
    __syncthreads(); // Ensure uni is loaded before proceeding

    unsigned int address = blockIdx.x * blockDim.x + baseIndex + threadIdx.x;

    if (address < n) // Boundary check for safety
    {
        g_data[address] += uni; // Perform addition at global memory address
    }
}