#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFillEmptySlots(short *cnewtri, int *cprefix, int *cempty, int *ctriangles, int nTris, int newnTris, int offset)
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (x >= nTris || cnewtri[x] < 0)
        return;

    __shared__ int sharedMemory[256]; // Utilize shared memory for temporary storage

    int value;

    if (x < newnTris)
        value = x;
    else {
        value = cempty[cprefix[x] - offset];

        int idx = threadIdx.x;
        for (int i = idx; i < 9; i += blockDim.x)
            sharedMemory[i] = ctriangles[x * 9 + i];  // Use shared memory for loading data

        __syncthreads();  // Synchronize threads to ensure all data is loaded

        for (int i = idx; i < 9; i += blockDim.x)
            ctriangles[value * 9 + i] = sharedMemory[i];  // Write data from shared memory to global memory
    }

    cprefix[x] = value;
}