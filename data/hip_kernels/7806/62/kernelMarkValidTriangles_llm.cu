#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelMarkValidTriangles(short *cnewtri, int *cvalid, int nTris)
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    // Early exit if the index exceeds number of triangles
    if (x >= nTris)
        return;

    // Check validity and mark the triangle in cvalid
    cvalid[x] = (cnewtri[x] >= 0) ? 1 : 0;
}