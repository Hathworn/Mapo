#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelCollectDeadTriangles(int *cdeadTri, short *cnewtri, int *cmarker, int nTris) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation for 1D grid

    if (x >= nTris || cnewtri[x] >= 0) // Boundary and condition check
        return;

    int id = cmarker[x];
    cdeadTri[id] = x; // Assign dead triangle index
}