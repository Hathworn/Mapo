#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFillEmptySlots1(short *cnewtri, int *cprefix, int *cempty, int *ctriangles, int nTris, int newnTris, int offset)
{
    int x = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (x >= nTris || cnewtri[x] < 0)
        return;

    int value;

    if (x < newnTris)
        value = x;
    else {
        value = cempty[cprefix[x] - offset]; // Determine empty slot index

        int base_src = x * 9; // Source index for ctriangles
        int base_dst = value * 9; // Destination index for ctriangles
        
        for (int i = 0; i < 9; i++) // Loop to copy the triangle
            ctriangles[base_dst + i] = ctriangles[base_src + i];
    }

    cprefix[x] = value; // Store the value in cprefix
}