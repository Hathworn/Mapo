#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelMissingDetection(int nVerts, int *cactive, int *cvertarr) {
    // Calculate unique global index for each thread
    int x = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Check for missing sites with boundary check in a single line
    if (x < nVerts) cactive[x] = (cvertarr[x] < 0) ? 0 : cactive[x];
}