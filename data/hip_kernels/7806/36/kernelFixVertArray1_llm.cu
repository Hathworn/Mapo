#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFixVertArray1(int *ctriangles, int nTris, int *cvertarr)
{
    // Calculate the unique thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Early return if thread index is out of bounds
    if (x >= nTris)
        return;

    // Load triangle vertices from global memory
    int v0 = ctriangles[x * 9 + 4];
    int v1 = ctriangles[x * 9 + 5];
    int v2 = ctriangles[x * 9 + 3];

    // Atomic exchange operations to update vertex array
    ctriangles[x * 9 + 6] = atomicExch(&cvertarr[v0], (x << 2));
    ctriangles[x * 9 + 7] = atomicExch(&cvertarr[v1], (x << 2) | 1);
    ctriangles[x * 9 + 8] = atomicExch(&cvertarr[v2], (x << 2) | 2);
}