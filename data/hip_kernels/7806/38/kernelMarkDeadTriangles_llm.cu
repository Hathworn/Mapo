#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelMarkDeadTriangles(int *cmarker, short *cnewtri, int nTris) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify calculation of thread index

    if (x < nTris) {  // Merge boundary check with assignment
        cmarker[x] = (cnewtri[x] >= 0 ? 0 : 1);
    }
}