#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelGenerateTriangles(int *voronoiPtr, short2 *patternPtr, int3 *ctriangles, int *offset, int width, int min, int max) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < min || x >= max)
        return;

    int xwidth = x * width;
    short2 pattern = patternPtr[xwidth + min];
    int i0, i1, i2, i3;
    int3 *pT = &ctriangles[offset[x - 1]];

    // Jump through all voronoi vertices in a texture row
    while (pattern.y > 0 && pattern.y < max) {
        i0 = voronoiPtr[xwidth + pattern.y];
        i1 = voronoiPtr[xwidth + pattern.y + 1];
        i2 = voronoiPtr[xwidth + width + pattern.y + 1];
        i3 = voronoiPtr[xwidth + width + pattern.y];

        // Use a switch case for better readability and performance
        switch (pattern.x) {
            case 0:
                *pT = make_int3(i3, i1, i2);
                break;
            case 1:
                *pT = make_int3(i0, i2, i3);
                break;
            case 2:
                *pT = make_int3(i1, i3, i0);
                break;
            case 3:
                *pT = make_int3(i2, i0, i1);
                break;
            case 4:
                // Generate 2 triangles
                *pT = make_int3(i2, i0, i1);
                pT++;
                *pT = make_int3(i3, i0, i2);
                break;
        }

        // Move to the next pattern
        pT++;
        pattern = patternPtr[xwidth + pattern.y + 1];
    }
}