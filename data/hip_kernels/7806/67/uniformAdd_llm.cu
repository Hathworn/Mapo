#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex)
{
    __shared__ int uni;
    
    // Load uniform value into shared memory
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];

    __syncthreads();

    unsigned int address = (blockIdx.x * blockDim.x * 2) + baseIndex + threadIdx.x;

    // Perform addition if within bounds
    if (address < n) {
        g_data[address] += uni;
    }

    if (address + blockDim.x < n) {
        g_data[address + blockDim.x] += uni;
    }
}