#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFixVertArray(int *ctriangles, int nTris, int *cvertarr)
{
    // Calculate unique global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Check bounds
    if (x >= nTris)
        return;

    // Store indices
    int idx = x * 9;

    // Extract vertices
    int v0 = ctriangles[idx + 4];
    int v1 = ctriangles[idx + 5];
    int v2 = ctriangles[idx + 3];

    // Perform atomic exchange operations
    ctriangles[idx + 6] = atomicExch(&cvertarr[v0], (x << 2));
    ctriangles[idx + 7] = atomicExch(&cvertarr[v1], (x << 2) | 1);
    ctriangles[idx + 8] = atomicExch(&cvertarr[v2], (x << 2) | 2);
}