#include "hip/hip_runtime.h"
```c++
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelMarkValidTriangles1(short *cnewtri, int *cvalid, int nTris)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    if (x < nTris) // Merge condition to reduce branching
    {
        cvalid[x] = (cnewtri[x] >= 0) ? 1 : 0;
    }
}