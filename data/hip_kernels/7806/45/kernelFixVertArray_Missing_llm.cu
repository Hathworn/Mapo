#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFixVertArray_Missing(int *ctriangles, int nTris, int *cvertarr)
{
    // Calculate global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (x >= nTris) return;

    // Load triangle vertices
    int v0 = ctriangles[x * 9 + 4];
    int v1 = ctriangles[x * 9 + 5];
    int v2 = ctriangles[x * 9 + 3];

    // Store vertex values using atomic exchange
    ctriangles[x * 9 + 6] = atomicExch(&cvertarr[v0], (x << 2));
    ctriangles[x * 9 + 7] = atomicExch(&cvertarr[v1], (x << 2) | 1);
    ctriangles[x * 9 + 8] = atomicExch(&cvertarr[v2], (x << 2) | 2);
}