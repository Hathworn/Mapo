#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function with an empty __syncthreads()
__global__ void sync_conv_groups() {
    __syncthreads();  // Synchronize all threads in the block
}