#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelMagicUpsampleX(float *in, int _w, int _h, float *out) {
    // Coefficients : 1/4, 3/4, 3/4, 1/4 in each direction (doubles the size of the picture)

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= _w || y >= _h)
        return;

    // Duplicate the points at the same place (?)
    out[y * 2 * _w + 2 * x] = in[y * _w + x];

    // Improved boundary checks and computation
    if (x < _w - 1) {
        float left = (x > 0) ? in[y * _w + x - 1] : 0.0f;
        float right = (x < _w - 2) ? in[y * _w + x + 2] : 0.0f;
        out[y * 2 * _w + 2 * x + 1] = __fdividef(3.0f * (in[y*_w + x] + in[y*_w + x + 1]) + left + right, 8.0f);
    }
}