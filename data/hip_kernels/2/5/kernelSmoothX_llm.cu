#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelSmoothX(float *in, int w, int h, float *out)
{
    // Calculate global thread indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure the thread is within bounds
    if (x >= w || y >= h) return;

    // Precompute row index
    int idx = y * w;

    // Calculate neighboring indices and clamp them
    int a = max(0, x - 2);
    int b = max(0, x - 1);
    int d = min(w - 1, x + 1);
    int e = min(w - 1, x + 2);

    // Perform the weighted sum to smooth
    out[idx + x] = 0.0625f * in[idx + a] + 0.25f * in[idx + b] + 0.375f * in[idx + x] + 0.25f * in[idx + d] + 0.0625f * in[idx + e];
}