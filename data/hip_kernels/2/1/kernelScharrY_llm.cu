#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelScharrY(float const *in, int _w, int _h, float *out)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure within bounds
    if (x >= _w || y >= _h)
        return;

    // Calculate row indices with boundary checks
    int a = max(y - 1, 0);
    int c = min(y + 1, _h - 1);

    // Calculate column indices with boundary checks
    int i1 = max(x - 1, 0);
    int i3 = min(x + 1, _w - 1);

    // Compute linear indices
    int a1 = a * _w + i1;
    int a2 = a * _w + x;
    int a3 = a * _w + i3;
    int c1 = c * _w + i1;
    int c2 = c * _w + x;
    int c3 = c * _w + i3;

    // Simplified arithmetic calculation
    float val = -in[a1] - in[a3] + in[c1] + in[c3];
    val = fmaf(10.0f, in[c2] - in[a2], val); // Fused multiply-add
    out[y * _w + x] = val * 0.05f; // Equivalent to division by 20.0
}