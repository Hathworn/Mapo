#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelScharrX( float const *in, int _w, int _h, float *out) {
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    if(x >= _w || y >= _h) return;

    int a = max(y-1, 0);
    int b = y;
    int c = min(y+1, _h -1);

    int i1 = max(x-1, 0);
    int i3 = min(x+1, _w-1);

    // Precompute repeated index calculations
    int a1 = a * _w + i1;
    int a3 = a * _w + i3;
    int b1 = b * _w + i1;
    int b3 = b * _w + i3;
    int c1 = c * _w + i1;
    int c3 = c * _w + i3;

    // Optimize calculation by avoiding repetitive computation
    float in_a1 = in[a1], in_c1 = in[c1], in_a3 = in[a3], in_c3 = in[c3], in_b3 = in[b3], in_b1 = in[b1];
    out[y * _w + x] = __fdividef(3.0f * (-in_a1 - in_c1 + in_a3 + in_c3) + 10.0f * (in_b3 - in_b1), 20.0f);
}