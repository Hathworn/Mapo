#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelSmoothY(float const * in, int w, int h, float * out)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Early exit if out of bounds
    if(x >= w || y >= h) return;

    // Use max/min to avoid repeated conditions
    int a = max(y - 2, 0);
    int b = max(y - 1, 0);
    int d = min(y + 1, h - 1);
    int e = min(y + 2, h - 1);

    // Optimize by storing the weights in shared memory if needed for larger data
    out[y * w + x] = 0.0625f * in[a * w + x] + 0.25f * in[b * w + x] + 
                     0.375f * in[y * w + x] + 0.25f * in[d * w + x] + 
                     0.0625f * in[e * w + x];
}