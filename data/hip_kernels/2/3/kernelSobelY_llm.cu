#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelSobelY(float const *in, int _w, int _h, float *out)
{
    // Acquire global indexes for thread
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Return if thread is out of bounds
    if(x >= _w || y >= _h)
        return;

    // Calculate clamped y positions to avoid boundary issues
    int a = max(y - 1, 0);
    int c = min(y + 1, _h - 1);

    // Calculate the clamped x positions to avoid boundary issues
    int i1 = max(x - 1, 0);
    int i3 = min(x + 1, _w - 1);

    // Calculate input array indexes based on clamped positions
    int a1 = a * _w + i1;
    int a2 = a * _w + x;
    int a3 = a * _w + i3;
    int c1 = c * _w + i1;
    int c2 = c * _w + x;
    int c3 = c * _w + i3;

    // Precompute values in a shared memory for reuse
    __shared__ float sharedInput[3][3];
    sharedInput[0][0] = in[a1];
    sharedInput[0][1] = in[a2];
    sharedInput[0][2] = in[a3];
    sharedInput[2][0] = in[c1];
    sharedInput[2][1] = in[c2];
    sharedInput[2][2] = in[c3];

    __syncthreads();

    // Reduce redundant computation
    float result = -1.0f * sharedInput[0][0] 
                 - 2.0f * sharedInput[0][1] 
                 - 1.0f * sharedInput[0][2] 
                 + 1.0f * sharedInput[2][0] 
                 + 2.0f * sharedInput[2][1] 
                 + 1.0f * sharedInput[2][2];

    // Write result back to global memory
    out[y * _w + x] = __fdividef(result, 4.0f);
}