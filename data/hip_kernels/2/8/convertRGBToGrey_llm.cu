#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Constant values on device
#define MAX_WEIGHT_VALUES 50
#define MIN_DET FLT_EPSILON

__constant__ __device__ int   LK_iteration;
__constant__ __device__ int   LK_patch;
__constant__ __device__ int   LK_points;
__constant__ __device__ int   LK_height;
__constant__ __device__ int   LK_width;
__constant__ __device__ int   LK_pyr_w;
__constant__ __device__ int   LK_pyr_h;
__constant__ __device__ int   LK_pyr_level;
__constant__ __device__ int   LK_width_offset;
__constant__ __device__ char  LK_init_guess;
__constant__ __device__ float LK_scaling;
__constant__ __device__ float LK_threshold;
__constant__ __device__ float LK_Weight[MAX_WEIGHT_VALUES];
__constant__ __device__ int   LK_win_size;

texture <float, 2, hipReadModeElementType> texRef_pyramid_prev;
texture <float, 2, hipReadModeElementType> texRef_pyramid_cur;
texture <float, 2, hipReadModeElementType> gpu_textr_pict_0;
texture <float, 2, hipReadModeElementType> gpu_textr_pict_1;
texture <float, 2, hipReadModeElementType> gpu_textr_deriv_x;
texture <float, 2, hipReadModeElementType> gpu_textr_deriv_y;

// Kernel to compute the tracking
__global__ void convertRGBToGrey(unsigned char *d_in, float *d_out, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index is within bounds
    if (idx < N)
    {
        // Use float constants with minimal decimal to improve precision
        float blue = d_in[idx * 3] * 0.1144f;
        float green = d_in[idx * 3 + 1] * 0.5867f;
        float red = d_in[idx * 3 + 2] * 0.2989f;

        // Reduce the computation in single line to optimize memory access
        d_out[idx] = blue + green + red;
    }
}