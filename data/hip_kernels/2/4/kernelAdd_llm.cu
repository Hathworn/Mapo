#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelAdd(float const *in1, float const *in2, int _w, int _h, float *out) {
    // Calculate global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within bounds
    if (x >= _w || y >= _h) return;

    // Pre-calculate index
    int index = y * _w + x;

    // Use HIP intrinsic functions for optimized arithmetic
    float v1 = in1[index];
    float v2 = in2[index];
    out[index] = __fsqrt_rn(__fadd_rn(__fmul_rn(v1, v1), __fmul_rn(v2, v2)));
}