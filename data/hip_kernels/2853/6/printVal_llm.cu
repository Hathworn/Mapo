#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void printVal(void) {
    // Use block and thread indices for better control parallel execution, even if not used currently
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Print a unique identifier for each thread to ensure correct mapping in future enhancements
    printf("Hola from thread %d!!\n", idx);
}