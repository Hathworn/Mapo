#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void k_zero_comp_xyz(float* __restrict__ data, uint n, uint stride)
{
    uint i = blockIdx.x * blockDim.x + threadIdx.x;
    uint p = blockIdx.y;
    
    // Ensure the thread index does not exceed bounds
    if (i >= n) return;

    // Use a pointer to avoid repeated calculations
    float* data_ptr = data + p * stride;
    
    // Zero the data at computed index
    data_ptr[i] = 0.f;
}