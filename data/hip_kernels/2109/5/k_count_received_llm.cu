#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory and unroll loop for better performance.
__global__ static void k_count_received(int nr_total_blocks, uint *d_n_recv_by_block, uint *d_spine_cnts)
{
    __shared__ uint shared_n_recv[THREADS_PER_BLOCK];

    int tid = threadIdx.x;
    int bid = tid + THREADS_PER_BLOCK * blockIdx.x;

    if (bid < nr_total_blocks) {
        // Read to shared memory to reduce global memory access.
        shared_n_recv[tid] = d_n_recv_by_block[bid];
        __syncthreads();

        // Unroll loop to enhance throughput.
        d_spine_cnts[bid * 10 + CUDA_BND_S_NEW] = shared_n_recv[tid];
    }
}