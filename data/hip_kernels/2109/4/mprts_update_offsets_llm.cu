#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void mprts_update_offsets(int nr_total_blocks, uint *d_off, uint *d_spine_sums)
{
    // Calculate the global index of the thread
    int bid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread is within bounds
    if (bid < nr_total_blocks) {
        // Load spine sums and store to d_off
        d_off[bid] = d_spine_sums[bid * CUDA_BND_STRIDE];
    }
}