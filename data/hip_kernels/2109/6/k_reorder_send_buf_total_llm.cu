#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void k_reorder_send_buf_total(int nr_prts, int nr_total_blocks, uint *d_bidx, uint *d_sums, float4 *d_xi4, float4 *d_pxi4, float4 *d_xchg_xi4, float4 *d_xchg_pxi4) 
{
    // Use variables to store block and thread indices for better readability
    int thread_id = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Process particles in a strided loop to improve occupancy and handle larger arrays
    for (int i = thread_id; i < nr_prts; i += stride) {
        if (d_bidx[i] == CUDA_BND_S_OOB) {
            int j = d_sums[i];
            d_xchg_xi4[j]  = d_xi4[i];
            d_xchg_pxi4[j] = d_pxi4[i];
        }
    }
}