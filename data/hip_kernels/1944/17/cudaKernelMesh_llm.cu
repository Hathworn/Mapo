#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaKernelMesh(float4* pos, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return; // Bounds check for valid indices

    // Calculate uv coordinates
    float u = 2.0f * (x / (float)width) - 1.0f;
    float v = 2.0f * (y / (float)height) - 1.0f;

    // Calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

    // Write output vertex
    pos[y * width + x] = make_float4(u, w, v, __int_as_float(0xff00ff00)); // Color : DirectX ARGB, OpenGL ABGR
}