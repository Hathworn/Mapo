#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaKernelTexture2D(unsigned char* surface, int width, int height, size_t pitch, float t)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check for thread bounds and return early if out of bounds
    if (x >= width || y >= height) return;

    unsigned char* pixel = (unsigned char*)(surface + y * pitch) + 4 * x;

    // Precompute trigonometric terms to optimize performance
    float cos_t = cos(t);
    float cos_tx = cos(t + 10.0f * ((2.0f * x) / width - 1.0f));
    float cos_ty = cos(t + 10.0f * ((2.0f * y) / height - 1.0f));

    // Populate pixel colors
    float value_x = 0.5f + 0.5f * cos_tx;
    float value_y = 0.5f + 0.5f * cos_ty;

    // Use precomputed cos_t
    pixel[0] = 255 * (0.5f + 0.5f * cos_t);                   // blue
    pixel[1] = 255 * (0.5f * pixel[1] / 255.0f + 0.5f * pow(value_y, 3.0f)); // green
    pixel[2] = 255 * (0.5f * pixel[0] / 255.0f + 0.5f * pow(value_x, 3.0f)); // red
    pixel[3] = 255;                                           // alpha
}
```
