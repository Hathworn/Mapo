#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
// Utilize shared memory to improve access speed
__shared__ float shared_outgrad[blockDim.x];
__shared__ float shared_indata[blockDim.x];

int tid = blockIdx.x * blockDim.x + threadIdx.x;
int local_tid = threadIdx.x;

// Load data into shared memory
if (tid < N) {
    shared_outgrad[local_tid] = outgrad[tid];
    shared_indata[local_tid] = indata[tid];
}
__syncthreads();

// Compute operation using data from shared memory
if (tid < N) {
    ingrad[tid] = shared_indata[local_tid] > 0 ? shared_outgrad[local_tid] : 0;
}
}