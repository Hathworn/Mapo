#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Calculate unique thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use grid-stride loop for better utilization
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        out[i] = fmaxf(in[i], 0.0f); // Use intrinsic function for better performance
    }
}