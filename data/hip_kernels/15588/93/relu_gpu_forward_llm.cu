#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  // Calculate the stride based on the number of threads

    // Use loop to increase occupancy and process more elements in one go
    for (; tid < N; tid += stride) {
        out[tid] = in[tid] > 0 ? in[tid] : 0;
    }
}