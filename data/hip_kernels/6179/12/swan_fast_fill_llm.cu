#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel to fill uint4 values
__global__ void swan_fast_fill( uint4 *ptr, int len ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Use loop unrolling and stride for better memory coalescing
    for (int i = idx; i < len; i += stride) {
        ptr[i] = make_uint4(0, 0, 0, 0);
    }
}