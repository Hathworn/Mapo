#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shmem(int *in, int *out, int N) {
    extern __shared__ int buf[];

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x; // Local index within the block

    // Load input into shared memory
    if (idx < N) {
        buf[tid] = in[idx];
    }
    __syncthreads();

    // Perform in-place reversal using shared memory
    if (tid < blockDim.x / 2) {
        int mirrorIdx = blockDim.x - tid - 1;
        if (idx + mirrorIdx < N) { // Ensure within bounds
            int tmp = buf[mirrorIdx];
            buf[mirrorIdx] = buf[tid];
            buf[tid] = tmp;
        }
    }
    __syncthreads();

    // Write result back to global memory
    if (idx < N) {
        out[idx] = buf[tid];
    }
}