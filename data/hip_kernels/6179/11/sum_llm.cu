#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel using vectorized operations
__global__ void sum(float4 *a, float4 *b, int N) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < N) {
        // Use vector addition directly
        a[idx].x += b[idx].x;
        a[idx].y += b[idx].y;
        a[idx].z += b[idx].z;
        a[idx].w += b[idx].w;
    }
}