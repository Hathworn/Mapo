#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float in1[1000];
__device__ __constant__ float in2[1000];

__global__ void vecadd(float* out, int N) {
    // Calculate the global index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure index is within bounds before accessing arrays
    if (idx < N) {
        out[idx] = in1[idx] + in2[idx];
    }
}

__global__ void vecadd(float* in1, float* in2, float* out, int N) {
    // Calculate the global index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure index is within bounds before accessing arrays
    if (idx < N) {
        out[idx] = in1[idx] + in2[idx];
    }
}