#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_2(float4 *localbuf, float4 *ptrd, int offset_0, int offset_1, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory for coalesced access
    extern __shared__ float4 shared[];
    
    if (idx < N) {
        shared[threadIdx.x] = ptrd[offset_0 + idx];
        float4 t2 = ptrd[offset_1 + idx];

        shared[threadIdx.x].x += t2.x;
        shared[threadIdx.x].y += t2.y;
        shared[threadIdx.x].z += t2.z;
        shared[threadIdx.x].w += t2.w;

        // Write back to global memory
        localbuf[idx] = shared[threadIdx.x];
    }
}