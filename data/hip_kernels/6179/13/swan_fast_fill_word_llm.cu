#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void swan_fast_fill_word(uint *ptr, int len) {
// Calculate global index and step size
int idx = threadIdx.x + blockDim.x * blockIdx.x;
int stride = gridDim.x * blockDim.x;

// Loop to fill memory in increments of stride
for (; idx < len; idx += stride) {
    ptr[idx] = 0;
}
}