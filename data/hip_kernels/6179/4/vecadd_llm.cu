#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float in1[1000];
__device__ __constant__ float in2[1000];

__global__ void vecadd(float *out, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        // Reduce global memory loads by using shared memory
        extern __shared__ float shared_in2[];
        if (threadIdx.x < 1000) {
            shared_in2[threadIdx.x] = in2[threadIdx.x];
        }
        __syncthreads();
        out[idx] = in1[idx] + shared_in2[idx];
    }
}

__global__ void vecadd(float *in1, float *in2, float *out, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        // Combine memory load and addition into one operation to reduce latency
        out[idx] = in1[idx] + in2[idx];
    }
}