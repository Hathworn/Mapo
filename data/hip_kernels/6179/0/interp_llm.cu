#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
texture<float, 1, hipReadModeElementType> tex_sin;
texture<float, 1, hipReadModeElementType> tex_sin2;
texture<float, 1, hipReadModeElementType> tex_sin3;

__global__ void interp(float* out, float *out2, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < N) {
        float normalizedIdx = (float)idx / N; // Precompute normalized index
        out[idx] = tex1D(tex_sin3, normalizedIdx);
        out2[idx] = tex1D(tex_sin, normalizedIdx);
    }
}