#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 256  // Define block size for better occupancy

__device__ float in1[1000];
__device__ __constant__ float in2[1000];

// Optimized kernel using shared memory
__global__ void vecadd_shared_memory(float* out, int N) {
    __shared__ float shared_in1[BLOCK_SIZE];
    __shared__ float shared_in2[BLOCK_SIZE];

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int threadIdxInBlock = threadIdx.x;

    // Load data into shared memory
    if (idx < N) {
        shared_in1[threadIdxInBlock] = in1[idx];
        shared_in2[threadIdxInBlock] = in2[idx];
    }
    __syncthreads();
    
    // Perform vector addition using shared memory
    if (idx < N) {
        out[idx] = shared_in1[threadIdxInBlock] + shared_in2[threadIdxInBlock];
    }
}

// Separately optimized kernel for external input arrays
__global__ void vecadd(float* in1, float* in2, float* out, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) {
        out[idx] = in1[idx] + in2[idx];
    }
}