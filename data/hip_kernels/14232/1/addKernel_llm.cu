#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global thread index to allow larger arrays
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check array bounds to prevent potential out-of-range access
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}
```
