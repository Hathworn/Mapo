#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int threads_per_block = 256;

__global__ void vector_max_kernel(float *in, float *out, int N) {
    int block_id = blockIdx.x + gridDim.x * blockIdx.y;
    int thread_id = blockDim.x * block_id + threadIdx.x;
    int tid = threadIdx.x;

    // Shared memory for maximum value in each block
    __shared__ float shared_max[threads_per_block];

    // Load input value into shared memory
    float local_max = (thread_id < N) ? in[thread_id] : -FLT_MAX;
    shared_max[tid] = local_max;
    __syncthreads();

    // Reduction to find maximum in the block, using shared memory
    for (int stride = threads_per_block / 2; stride > 0; stride >>= 1) {
        if (tid < stride && thread_id + stride < N) {
            shared_max[tid] = fmaxf(shared_max[tid], shared_max[tid + stride]);
        }
        __syncthreads();
    }

    // Write the block's maximum to the output
    if (tid == 0) {
        out[block_id] = shared_max[0];
    }
}