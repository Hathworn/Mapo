#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateGradientsKernel(float4 *D, float4 *TD, unsigned int nVertices)
{
    int baseIndex = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    
    // Unroll the loop for better performance
    #pragma unroll
    for (int offset = 0; offset < 4 * BLOCK_SIZE_AVGG; offset += BLOCK_SIZE_AVGG)
    {
        D[baseIndex + offset] = TD[baseIndex + offset];
    }
}