#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Reorganized kernel to optimize memory access and reduce redundant computation
__global__ void computeSphereVertexDistancesKernel(float4 *V, float *dist, unsigned int *NEIGHBOR, unsigned int *NBOFFSETS, unsigned int *nNeighbors, unsigned int nVertices, float circumference)
{
    int n, N;
    int offset, soffset;

    // Calculate global index and base vertex index
    int vidxb = 4 * (blockIdx.x * blockDim.x) + threadIdx.x;
    int basevert = 4 * (blockIdx.x * blockDim.x);

    int bidx = threadIdx.x;
    
    // Shared memory for caching vertex positions
    __shared__ float4 SI[4 * BLOCK_SIZE_CVD];

    // Load vertices into shared memory
    for (int vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_CVD; vidx += BLOCK_SIZE_CVD)
    {
        if (vidx < nVertices)
        {
            SI[bidx] = V[vidx];
            bidx += BLOCK_SIZE_CVD;
        }
    }
    __syncthreads();

    bidx = threadIdx.x;

    // Process vertices and compute distances
    for (int vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_CVD; vidx += BLOCK_SIZE_CVD)
    {
        if (vidx < nVertices)
        {
            offset = NBOFFSETS[vidx];
            N = nNeighbors[vidx];
            float4 tv = SI[bidx];

            bidx += BLOCK_SIZE_CVD;

            for (n = 0; n < N; n++)
            {
                soffset = NEIGHBOR[offset + n];
                int tab = soffset - basevert;
                float4 nv = (tab > 0 && tab < 4 * BLOCK_SIZE_CVD) ? SI[tab] : V[soffset];

                float dot = __fmaf_rn(tv.x, nv.x, __fmaf_rn(tv.y, nv.y, __fmul_rn(tv.z, nv.z)));
                float n1 = __fmaf_rn(tv.x, tv.x, __fmaf_rn(tv.y, tv.y, __fmul_rn(tv.z, tv.z)));
                float n2 = __fmaf_rn(nv.x, nv.x, __fmaf_rn(nv.y, nv.y, __fmul_rn(nv.z, nv.z)));
                float norm = __sqrtf(n1) * __sqrtf(n2);

                if (norm < 1.0e-7f || fabsf(dot) > norm)
                {
                    dist[offset + n] = 0.0f;
                }
                else
                {
                    dist[offset + n] = circumference * fabsf(acosf(dot / norm));
                }
            }
        }
    }
}