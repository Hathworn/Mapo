#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GradientAverageKernel(float4 *D, float4 *TD, unsigned int *NEIGHBOR, unsigned int *NBOFFSETS, unsigned int *nNeighbors, unsigned int nVertices)
{
    int vidxb = 4 * (blockIdx.x * blockDim.x) + threadIdx.x;
    if (vidxb >= nVertices) return;  // Early exit if out of bounds

    __shared__ float4 SI[4 * BLOCK_SIZE_AVGG];

    int bidx = 4 * threadIdx.x;
    // Load into shared memory
    for (int vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_AVGG && vidx < nVertices; vidx += BLOCK_SIZE_AVGG)
    {
        SI[bidx] = D[vidx];
        bidx++;
    }
    
    __syncthreads();  // Ensure shared memory is fully populated before use

    bidx = 4 * threadIdx.x;
    // Main computation
    for (int vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_AVGG && vidx < nVertices; vidx += BLOCK_SIZE_AVGG)
    {
        int offset = NBOFFSETS[vidx];
        int N = nNeighbors[vidx];
        float4 td = SI[bidx++];

        for (int n = 0; n < N; n++)
        {
            int soffset = NEIGHBOR[offset + n];
            float4 nbd = D[soffset];
            td.x += nbd.x;
            td.y += nbd.y;
            td.z += nbd.z;
        }

        // Averaging the neighbors
        float invN = 1.0f / (N + 1);
        td.x *= invN;
        td.y *= invN;
        td.z *= invN;

        TD[vidx] = td;
    }
}