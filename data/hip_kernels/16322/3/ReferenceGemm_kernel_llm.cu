#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ReferenceGemm_kernel(int M, int N, int K, float alpha, float const *A, int lda, float const *B, int ldb, float beta, float *C, int ldc) {

    // Calculate the row index of the C element and the shared row of A
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // Calculate the column index of C and the shared column of B
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if (i < M && j < N) {
        float accumulator = 0.0f;
        
        for (int k = 0; k < K; ++k) {
            // Use registers to prevent re-computation
            float aElement = A[i + k * lda];     
            float bElement = B[k + j * ldb];
            accumulator += aElement * bElement;
        }

        // Perform final computation for C
        C[i + j * ldc] = alpha * accumulator + beta * C[i + j * ldc];
    }
}