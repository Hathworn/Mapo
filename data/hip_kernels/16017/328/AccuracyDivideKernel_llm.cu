#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Global kernel function to divide accuracy by N
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Each thread calculates its portion of the division
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx == 0) { // Only the first thread performs the division
        *accuracy /= N;
    }
}