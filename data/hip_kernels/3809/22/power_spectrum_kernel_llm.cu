#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;

    // Calculate values for each thread
    __shared__ float shared_A[CU1DBLOCK * 2]; // Shared memory for improved access

    if (thread_id < half_length) {
        float2 val = reinterpret_cast<float2 *>(Ar)[thread_id];
        shared_A[thread_id] = val.x * val.x + val.y * val.y;
    }
    __syncthreads(); // Ensure all threads have computed their results

    // Write results back to global memory
    for (int idx = thread_id; idx < half_length; idx += CU1DBLOCK) {
        Aw[idx] = shared_A[idx];
    }

    // Handle special case using thread 0
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];
        Aw[0] = real * real;
        Aw[half_length] = im * im;
    }
}