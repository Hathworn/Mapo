#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplicarMatrices(float *m1, float *m2, float *mr, int columna1, int fila1, int columna2, int fila2)
{
    // Compute global row and column index
    int fila_r = blockIdx.y * blockDim.y + threadIdx.y;
    int columna_r = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize result accumulator
    float tmp_mult = 0;

    // Ensure indices are within bounds
    if (fila_r < fila2 && columna_r < columna1) {

        // Use shared memory to reduce global memory accesses
        __shared__ float shared_m1[/* appropriate size based on block */];
        __shared__ float shared_m2[/* appropriate size based on block */];

        // Load each stripe of both matrices into shared memory
        for (int i = 0; i < fila2; i += blockDim.y) {
            if (fila_r + i < fila2) {
                shared_m1[threadIdx.y] = m1[i + columna1 * (fila_r + i)];
                shared_m2[threadIdx.x] = m2[columna_r + columna2 * threadIdx.y];
            }
            __syncthreads();
            
            // Compute partial result using shared memory
            for (int j = 0; j < blockDim.y; ++j) {
                tmp_mult += shared_m1[j] * shared_m2[j];
            }
            __syncthreads();
        }

        // Store accumulated result to output matrix
        mr[fila_r * columna2 + columna_r] = tmp_mult;
    }
}