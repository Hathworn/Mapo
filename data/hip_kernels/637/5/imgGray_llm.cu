#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void imgGray(unsigned char *d_image, unsigned char *d_imagegray, int width, int height) {
    // Compute single index to improve readability and reuse
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((width > col) && (height > row)) {
        int index = row * width + col;
        int rgbIndex = index * 3;
        // Use a float for coefficients to avoid additional casting
        d_imagegray[index] = d_image[rgbIndex + 2] * 0.299f 
                           + d_image[rgbIndex + 1] * 0.587f 
                           + d_image[rgbIndex] * 0.114f;
    }
}