#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void imgGray(unsigned char * d_image, unsigned char* d_imagegray, int width, int height) {
    // Calculate row and col using block and thread indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if indices are within image bounds
    if (col < width && row < height) {
        int idx = row * width + col; // Calculate 1D linear index for the pixel
        int rgb_idx = idx * 3; // Calculate 1D index for the RGB components
        // Convert RGB to grayscale using weighted average
        d_imagegray[idx] = d_image[rgb_idx + 2] * 0.299f + d_image[rgb_idx + 1] * 0.587f + d_image[rgb_idx] * 0.114f;
    }
}