#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Copy_Optimized_double(double *a, double *b, size_t len)
{
    // Calculate global thread index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Use strided loop for improved memory access pattern
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x) {
        b[i] = a[i];
    }
}