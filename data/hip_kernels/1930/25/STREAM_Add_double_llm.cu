#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Add_double(double *a, double *b, double *c, size_t len)
{
    // Use a single loop iteration to reduce overhead.
    // Check if index is within bounds before computation.
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len) {
        c[idx] = a[idx] + b[idx];
    }
}