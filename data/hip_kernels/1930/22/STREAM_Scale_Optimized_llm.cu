#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Scale_Optimized(float *a, float *b, float scale, size_t len)
{
    // Calculate global thread index and avoid out-of-bounds using early return
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx >= len) return;

    // Perform scaling operation
    b[idx] = scale * a[idx];
}