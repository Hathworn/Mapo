#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Copy_Optimized(float *a, float *b, size_t len)
{
    // Calculate global thread index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Loop over the elements with stride of total threads
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x)
    {
        b[i] = a[i];
    }
}