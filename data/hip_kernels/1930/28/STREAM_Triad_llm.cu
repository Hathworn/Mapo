#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Triad(float *a, float *b, float *c, float scalar, size_t len)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x; // Precompute the stride for better performance
    for (size_t i = idx; i < len; i += stride) { // Use a for loop for better readability
        c[i] = a[i] + scalar * b[i]; // Optimize memory access by computing within the loop
    }
}