#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Add_Optimized(float *a, float *b, float *c, size_t len)
{
    // Calculate unique global index for each thread
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Use a stride loop to ensure all elements are processed by threads
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x)
    {
        c[i] = a[i] + b[i];
    }
}