#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_array_double(double *a, double value, size_t len)
{
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Ensure loop is only executed if idx is within bounds initially
    if (idx < len) {
        for (; idx < len; idx += blockDim.x * gridDim.x) {
            a[idx] = value;  // Set value directly
        }
    }
}