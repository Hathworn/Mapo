#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Copy_double(double *a, double *b, size_t len)
{
    // Use the loop unrolling technique to minimize loop overhead
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < len)
    {
        b[idx] = a[idx];
    }
    idx += blockDim.x * gridDim.x;
    if (idx < len)
    {
        b[idx] = a[idx];
    }
    idx += blockDim.x * gridDim.x;
    if (idx < len)
    {
        b[idx] = a[idx];
    }
    idx += blockDim.x * gridDim.x;
    if (idx < len)
    {
        b[idx] = a[idx];
    }
    // Process remaining elements if any
    idx += blockDim.x * gridDim.x;
    while (idx < len)
    {
        b[idx] = a[idx];
        idx += blockDim.x * gridDim.x;
    }
}