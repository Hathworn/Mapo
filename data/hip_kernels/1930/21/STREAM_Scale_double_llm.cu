#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Scale_double(double *a, double *b, double scale, size_t len)
{
    // Calculate global thread index
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Using a stride loop to ensure all elements are processed
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x) {
        b[i] = scale * a[i];
    }
}