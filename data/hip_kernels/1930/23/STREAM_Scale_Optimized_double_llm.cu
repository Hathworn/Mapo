#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void STREAM_Scale_Optimized_double(double *a, double *b, double scale, size_t len)
{
    // Calculate global index for each thread
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Iterate over all elements by striding with total number of threads
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x) {
        b[i] = scale * a[i];
    }
}