#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Triad_Optimized_double(double *a, double *b, double *c, double scalar, size_t len)
{
    // Calculate global index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure work only if idx is within bounds
    if (idx < len) {
        c[idx] = a[idx] + scalar * b[idx];
    }
}