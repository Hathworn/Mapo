#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Triad_Optimized(float *a, float *b, float *c, float scalar, size_t len)
{
    // Use grid stride loop for optimal out-of-bound checks
    for (size_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < len; idx += blockDim.x * gridDim.x) 
    {
        c[idx] = a[idx] + scalar * b[idx];
    }
}