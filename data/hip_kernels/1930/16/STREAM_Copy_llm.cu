#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Copy(float *a, float *b, size_t len)
{
    // Optimize by using a more efficient block-stride loop with grid-stride
    for (size_t idx = threadIdx.x + blockIdx.x * blockDim.x; idx < len; idx += blockDim.x * gridDim.x) {
        b[idx] = a[idx];
    }
}