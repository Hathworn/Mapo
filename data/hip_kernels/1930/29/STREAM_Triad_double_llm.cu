#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Triad_double(double *a, double *b, double *c, double scalar, size_t len)
{
    // Optimize indexing by calculating once outside the loop
    size_t stride = blockDim.x * gridDim.x;
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use grid-stride loop for better performance
    for (; idx < len; idx += stride) {
        c[idx] = a[idx] + scalar * b[idx];
    }
}