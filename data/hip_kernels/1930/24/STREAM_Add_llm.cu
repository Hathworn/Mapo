#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Add(float *a, float *b, float *c, size_t len)
{
    // Calculate global index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Efficient loop limiting with stride
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x)
    {
        c[i] = a[i] + b[i];
    }
}