#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Scale(float *a, float *b, float scale, size_t len)
{
    // Calculate global index
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize loop using a grid-stride loop
    for (; idx < len; idx += blockDim.x * gridDim.x) {
        b[idx] = scale * a[idx];
    }
}