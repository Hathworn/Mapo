#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void STREAM_Add_Optimized_double(double *a, double *b, double *c, size_t len)
{
    // Calculate global thread index
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Process all elements by iterating over the grid
    for (size_t i = idx; i < len; i += blockDim.x * gridDim.x)
    {
        c[i] = a[i] + b[i];
    }
}