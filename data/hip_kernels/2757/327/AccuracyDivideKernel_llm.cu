#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory to reduce global memory accesses
    __shared__ float shared_accuracy;

    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy;
    }
    
    __syncthreads();
    
    // Perform division in shared memory; ensures only one division operation
    if (threadIdx.x == 0) {
        shared_accuracy /= N;
        *accuracy = shared_accuracy;
    }
}