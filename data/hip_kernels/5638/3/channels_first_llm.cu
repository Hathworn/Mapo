#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void channels_first(float* input, float* rinput, int channels, int height, int width, int pad_size)
{
    // Calculate indices
    int n = blockIdx.x;
    int y = blockIdx.y;
    int x = blockIdx.z;
    int ch_off = threadIdx.x;

    // Precompute dimensions
    int dimcyx = channels * height * width;
    int dimyx = height * width;
    int p_dimx = (width + 2 * pad_size);
    int p_dimy = (height + 2 * pad_size);
    int p_dimyxc = channels * p_dimy * p_dimx;
    int p_dimxc = p_dimx * channels;

    // Use shared memory to reduce global memory access
    __shared__ float smem[THREADS_PER_BLOCK];

    for (int c = ch_off; c < channels; c += THREADS_PER_BLOCK) {
        smem[threadIdx.x] = input[n * dimcyx + c * dimyx + y * width + x];  // Load into shared memory
        __syncthreads(); // Synchronize to ensure all threads have written to shared memory 

        rinput[n * p_dimyxc + (y + pad_size) * p_dimxc + (x + pad_size) * channels + c] = smem[threadIdx.x];  // Store from shared memory to output
        __syncthreads(); // Ensure the shared memory copy is complete before the next iteration
    }
}