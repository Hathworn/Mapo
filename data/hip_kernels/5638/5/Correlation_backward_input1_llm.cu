#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Correlation_backward_input1(int item, float *gradInput1, int nInputChannels, int inputHeight, int inputWidth, float *gradOutput, int nOutputChannels, int outputHeight, int outputWidth, float *rInput2, int pad_size, int kernel_size, int max_displacement, int stride1, int stride2)
{
    // Calculate global coordinates
    int n = item;
    int y = blockIdx.x * stride1 + pad_size;
    int x = blockIdx.y * stride1 + pad_size;
    int c = blockIdx.z;
    
    // Calculate shared memory index
    int tch_off = threadIdx.x;

    // Precompute constants
    int kernel_rad = (kernel_size - 1) / 2;
    int displacement_rad = max_displacement / stride2;
    int displacement_size = 2 * displacement_rad + 1;
    int pInputWidth = inputWidth + 2 * pad_size;

    // Compute bounds for iteration
    int xmin = max((x - kernel_rad - max_displacement) / stride1, 0);
    int ymin = max((y - kernel_rad - max_displacement) / stride1, 0);
    int xmax = min((x + kernel_rad - max_displacement) / stride1, outputWidth - 1);
    int ymax = min((y + kernel_rad - max_displacement) / stride1, outputHeight - 1);

    if (xmax < 0 || ymax < 0 || xmin > xmax || ymin > ymax) {
        return;
    }

    // Precompute offsets
    int pdimyxc = pInputWidth * nInputChannels * (inputHeight + 2 * pad_size);
    int pdimxc = pInputWidth * nInputChannels;
    int pdimc = nInputChannels;

    int tdimyx = outputHeight * outputWidth;
    int tdimx = outputWidth;

    int odimcyx = nInputChannels * inputHeight * inputWidth;
    int odimyx = inputHeight * inputWidth;
    int odimx = inputWidth;

    float nelems = kernel_size * kernel_size * nInputChannels;

    // Use shared memory for keeping accumulated products
    __shared__ float prod_sum[CUDA_NUM_THREADS];
    prod_sum[tch_off] = 0.0f;

    // Loop over output channels
    for (int tc = tch_off; tc < nOutputChannels; tc += CUDA_NUM_THREADS) {
        int i2 = (tc % displacement_size - displacement_rad) * stride2;
        int j2 = (tc / displacement_size - displacement_rad) * stride2;

        int indx2 = n * pdimyxc + (y + j2) * pdimxc + (x + i2) * pdimc + c;

        float val2 = rInput2[indx2];

        // Accumulate product within bounds
        for (int j = ymin; j <= ymax; ++j) {
            for (int i = xmin; i <= xmax; ++i) {
                int tindx = n * nOutputChannels * tdimyx + tc * tdimyx + j * tdimx + i;
                prod_sum[tch_off] += gradOutput[tindx] * val2;
            }
        }
    }
    __syncthreads();

    // Reduce the sum across threads
    if (tch_off == 0) {
        float reduce_sum = 0.0f;
        for (int idx = 0; idx < CUDA_NUM_THREADS; idx++) {
            reduce_sum += prod_sum[idx];
        }
        const int indx1 = n * odimcyx + c * odimyx + (y - pad_size) * odimx + (x - pad_size);
        gradInput1[indx1] = reduce_sum / nelems;
    }
}