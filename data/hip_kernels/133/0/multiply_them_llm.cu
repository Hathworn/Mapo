#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiply_them(float *dest, float *a, float *b)
{
    // Calculate global thread index for larger data handling
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Safety check: Ensure we don't exceed array bounds
    if (idx < N)  // N should be the size of the arrays
    {
        dest[idx] = a[idx] * b[idx];
    }
}
```
