#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void concat_z(size_t sz, float_t* src, float_t* dest, float_t* z, size_t stride)
{
    // Use shared memory to avoid redundant global accesses
    extern __shared__ float_t shared_mem[];
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < sz)
    {
        // Load into shared memory
        if (index >= stride)
        {
            shared_mem[index] = src[index - stride];
        }
        else
        {
            shared_mem[index] = z[index];
        }
        // Write back from shared memory to global memory
        dest[index] = shared_mem[index];
    }
}