#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose(size_t sz, float_t* src, float_t* dest, size_t src_width, size_t src_height)
{
    // Calculate the index using block and thread indices for parallel execution
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;

    if(index < sz)
    {
        // Use integer division and modulus for row-column conversion
        size_t i = index / src_width;
        size_t j = index % src_width;

        // Compute the destination index for transposed matrix
        size_t dest_index = j * src_height + i;
        
        // Assign value in transposed position
        dest[dest_index] = src[index];
    }
}