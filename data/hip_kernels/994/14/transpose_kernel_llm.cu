#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_kernel(size_t sz, float_t* src, float_t* dest, size_t ld_src, size_t ld_dest)
{
    size_t xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    size_t yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    size_t src_index = yIndex * ld_src + xIndex;
    size_t dest_index = xIndex * ld_dest + yIndex;

    // Check if the source index is within bounds
    if (src_index < sz) 
    {
        dest[dest_index] = src[src_index]; // Perform the transpose
    }
}