#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transformation(size_t num_values, float_t* src, float_t* dest, size_t ld_src, size_t ld_dest)
{
    // Calculate global thread index
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (index < num_values)
    {
        // Precompute reused values for optimization
        size_t block_idx = index / ld_src;
        size_t block_offset = (index % ld_src);
        size_t sub_block = block_offset % 8;
        size_t sub_offset = block_offset / 8;

        // Compute destination index
        size_t dest_index = block_idx * ld_src + sub_block * ld_dest + sub_offset;
        
        // Perform transformation
        dest[dest_index] = src[index];
    }
}