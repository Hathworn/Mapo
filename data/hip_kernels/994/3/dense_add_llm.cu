#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dense_add(size_t sz, float_t* src, float_t* dest)
{
    // Use shared memory for better access speed
    extern __shared__ float_t sharedSrc[];
    
    // Each block loads its portion of src into shared memory
    if(threadIdx.x < sz)
    {
        sharedSrc[threadIdx.x] = src[threadIdx.x];
    }
    __syncthreads();

    // Compute global index for dest
    size_t destIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only add if within bounds, using shared memory for src
    if(destIndex < sz && threadIdx.x < sz)
    {
        dest[destIndex] += sharedSrc[threadIdx.x];
    }
}