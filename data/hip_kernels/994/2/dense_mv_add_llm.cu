#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void dense_mv_add(size_t sz, float_t* src, float_t* dest)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < sz)
    {
        // Coalesced memory access
        dest[index] += src[index];
    }
}