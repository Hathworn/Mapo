#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reshape(size_t num_values, float_t* src, float_t* dest, size_t ld_src, size_t ld_dest)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < num_values)
    {
        // Improve memory access pattern by reducing computation inside.
        size_t row = index / ld_dest;
        size_t col = index % ld_dest;
        size_t src_index = row * ld_src + col;
        dest[index] = src[src_index];
    }
}