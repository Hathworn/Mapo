#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dense_add_conv(size_t sz, float_t* src, float_t* dest, size_t bias_dim)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t src_index = index % bias_dim; // Use modulus to obtain correct src index
    if (index < sz)
    {
        dest[index] += src[src_index]; // Access src using calculated src_index
    }
}