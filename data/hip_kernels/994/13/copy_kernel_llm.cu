#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(size_t sz, float_t* src, float_t* dest)
{
    // Calculate the global index
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure multiple elements processed per thread where possible
    size_t stride = gridDim.x * blockDim.x;

    // Use stride loop to allow for coalesced memory access and better utilization
    for (size_t i = index; i < sz; i += stride)
    {
        dest[i] = src[i];
    }
}