#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void affine_transform(size_t sz, float_t* audio, float_t* end_out, size_t stride)
{
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimized computation:
    if (index < sz)
    {
        float_t end_out_val = end_out[index];
        float_t exp_val = __expf(end_out[index + stride]); // Use __expf for faster computation
        audio[index + stride] = (audio[index + stride] - end_out_val) / exp_val;
    }
}