#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access and occupancy
__global__ void skip_add(size_t sz, const float_t* __restrict__ f1, float* __restrict__ skip_out_sum)
{
    // Calculate unique thread index
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary condition and perform the addition
    if(index < sz)
    {
        skip_out_sum[index] += f1[index];
    }
}