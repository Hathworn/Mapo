#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dropout_op(size_t sz, float_t* random_nums, float_t* data, float_t drop_rate, float_t scale) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check index boundaries once at the start
    if (index >= sz) return;
    
    // Simplify branching logic
    data[index] *= (random_nums[index] > drop_rate) * scale;
}