#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void skip_res_add(size_t sz, float_t* f5, float* f1, float_t* skip_out_sum, size_t stride)
{
    size_t index = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory for faster data access
    extern __shared__ float_t shared_mem[];

    if(index < sz)
    {
        // Load data from global to shared memory
        shared_mem[threadIdx.x] = f5[index + stride];
        
        // Sync threads to ensure all data is loaded
        __syncthreads();

        skip_out_sum[index] += shared_mem[threadIdx.x];
        f1[index] += f5[index];
    }
}