#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel with optimized memory accesses
__global__ void pw_copy_rc_cu_z(const double * __restrict__ din, double * __restrict__ zout, const int n) {
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Utilize shared memory for coalesced writing, improve cache usage
    __shared__ double shared_zout[256]; // Example size, should be set based on occupancy considerations

    if (igpt < n) {
        shared_zout[threadIdx.x * 2] = din[igpt];
        shared_zout[threadIdx.x * 2 + 1] = 0.0e0;
        
        __syncthreads(); // Synchronizing threads within block to ensure all writes to shared memory are done

        // Write back to global memory
        zout[2 * igpt] = shared_zout[threadIdx.x * 2];
        zout[2 * igpt + 1] = shared_zout[threadIdx.x * 2 + 1];
    }
}