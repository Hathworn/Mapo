#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_copy_cr_cu_z(const double *zin, double *dout, const int n) {
    // Calculate global thread ID using a 1-D grid configuration
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if this thread should perform a copy
    if (igpt < n) {
        dout[igpt] = zin[2 * igpt]; // Perform the copy
    }
}