#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_scatter_cu_z(double *c, const double *pwcc, const double scale, const int ngpts, const int nmaps, const int *ghatmap) {

    // Calculate global index using blockIdx and threadIdx
    const int igpt = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    // Ensure the thread operates within valid bounds
    if (igpt < ngpts) {
        int gmapIdx = ghatmap[igpt];
        int offset = 2 * igpt;

        // Optimize memory access by reusing computed indices
        double pwcc_real = scale * pwcc[offset];
        double pwcc_imag = scale * pwcc[offset + 1];

        c[2 * gmapIdx] = pwcc_real;
        c[2 * gmapIdx + 1] = pwcc_imag;

        // Conditional map processing
        if (nmaps == 2) {
            gmapIdx = ghatmap[igpt + ngpts];
            c[2 * gmapIdx] = pwcc_real;
            c[2 * gmapIdx + 1] = -pwcc_imag;
        }
    }
}