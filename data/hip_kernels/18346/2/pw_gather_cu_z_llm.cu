#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_gather_cu_z(double *pwcc, const double *c, const double scale, const int ngpts, const int *ghatmap) {
    const int igpt = 
    (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;

    // Optimize by adjusting the condition to eliminate unnecessary operations
    if (igpt >= ngpts) return;
    
    // Optimize memory access pattern for coalescing
    int index = 2 * ghatmap[igpt];
    pwcc[2 * igpt] = scale * c[index];
    pwcc[2 * igpt + 1] = scale * c[index + 1];
}