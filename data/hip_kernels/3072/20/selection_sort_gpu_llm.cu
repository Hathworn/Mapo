#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, float *dist, int *idx, float *val) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    idx += m * k * batch_index;
    val += m * k * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    float *p_dist;
    // Loop unrolling for improved performance
    for (int j = index; j + 4 <= m; j += stride * 4) {
        for (int offset = 0; offset < 4; ++offset) {
            int current_j = j + offset;
            if (current_j < m) {
                p_dist = dist + current_j * n;
                for (int s = 0; s < k; ++s) {
                    int min = s;
                    for (int t = s + 1; t < n; ++t) {
                        if (p_dist[t] < p_dist[min]) {
                            min = t;
                        }
                    }
                    idx[current_j * n + s] = min;
                    val[current_j * n + s] = p_dist[min];
                    float tmp = p_dist[min];
                    p_dist[min] = p_dist[s];
                    p_dist[s] = tmp;
                }
            }
        }
    }
    // Handle remaining elements when m is not a multiple of 4
    for (int j = index + (m / stride) * stride; j < m; j += stride) {
        p_dist = dist + j * n;
        for (int s = 0; s < k; ++s) {
            int min = s;
            for (int t = s + 1; t < n; ++t) {
                if (p_dist[t] < p_dist[min]) {
                    min = t;
                }
            }
            idx[j * n + s] = min;
            val[j * n + s] = p_dist[min];
            float tmp = p_dist[min];
            p_dist[min] = p_dist[s];
            p_dist[s] = tmp;
        }
    }
}