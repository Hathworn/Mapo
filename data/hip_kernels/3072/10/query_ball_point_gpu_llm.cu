#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    // Use blockIdx, blockDim, and threadIdx for parallel indexing
    int bs_index = blockIdx.x; // batch size index
    int pt_index = blockIdx.y * blockDim.x + threadIdx.x; // point index for xyz2

    if (bs_index < b && pt_index < m) {
        int cnt = 0;
        float x2 = xyz2[pt_index * 3 + 0];
        float y2 = xyz2[pt_index * 3 + 1];
        float z2 = xyz2[pt_index * 3 + 2];

        for (int k = 0; k < n; ++k) {
            if (cnt == nsample) break; // Stop if nsample points are found

            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            float d = max(sqrtf((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1)), 1e-20f);

            if (d < radius) {
                if (cnt == 0) {
                    // Initialize indices to k for valid repeating indices
                    for (int l = 0; l < nsample; ++l)
                        idx[pt_index * nsample + l] = k;
                }
                idx[pt_index * nsample + cnt] = k;
                cnt += 1;
            }
        }
    }
}