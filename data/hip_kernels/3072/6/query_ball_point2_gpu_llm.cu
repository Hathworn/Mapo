#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point2_gpu(int b, int n, int m, int nsample, const float *xyz1, const float *xyz2, const float *radii, int *idx, int *pts_cnt) {
    int batch_index = blockIdx.x;
    xyz1 += n * 3 * batch_index;
    xyz2 += m * 3 * batch_index;
    radii += m * batch_index;
    idx += m * nsample * batch_index;
    pts_cnt += m * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int j = index; j < m; j += stride) {
        int cnt = 0;
        float x2 = xyz2[j * 3 + 0];
        float y2 = xyz2[j * 3 + 1];
        float z2 = xyz2[j * 3 + 2];
        float radius = radii[j];

        for (int k = 0; k < n && cnt < nsample; ++k) {
            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];

            // Simplified distance calculation and comparison
            float dx = x2 - x1;
            float dy = y2 - y1;
            float dz = z2 - z1;
            float d2 = dx * dx + dy * dy + dz * dz;

            if (d2 < radius * radius) {
                if (cnt == 0) {
                    for (int l = 0; l < nsample; ++l) {
                        idx[j * nsample + l] = -1;
                    }
                }
                idx[j * nsample + cnt] = k;
                cnt += 1;
            }
        }
        pts_cnt[j] = cnt;
    }
}