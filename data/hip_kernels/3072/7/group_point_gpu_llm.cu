#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n * c * batch_index;
    idx += m * nsample * batch_index;
    out += m * nsample * c * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Unroll loop j for better parallel execution and memory access optimization
    for (int j = index; j < m; j += stride) {
        int base_out_idx = j * nsample * c;
        int base_idx = j * nsample;
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[base_idx + k];
            int base_points_idx = ii * c;
            int base_out_sample_idx = base_out_idx + k * c;
            for (int l = 0; l < c; ++l) {
                out[base_out_sample_idx + l] = points[base_points_idx + l];
            }
        }
    }
}