#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n * c * batch_index;
    idx += m * nsample * batch_index;
    out += m * nsample * c * batch_index;
    
    int index = threadIdx.x + blockIdx.y * blockDim.x;  // Use 2D grid for better mapping
    int stride = blockDim.x * gridDim.y;  // Adjust stride based on total threads
    
    // Unroll loops for better performance
    for (int j = index; j < m; j += stride) {
        #pragma unroll
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            #pragma unroll
            for (int l = 0; l < c; ++l) {
                out[j * nsample * c + k * c + l] = points[ii * c + l];
            }
        }
    }
}