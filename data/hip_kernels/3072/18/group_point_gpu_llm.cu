#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch = blockIdx.x; // Use blockIdx for each batch
    int thread = threadIdx.x; // Use threadIdx for parallel execution among threads
    points += n * c * batch;
    idx += m * nsample * batch;
    out += m * nsample * c * batch;

    for (int j = thread; j < m; j += blockDim.x) { // Distribute work using thread index
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            for (int l = 0; l < c; ++l) {
                out[j * nsample * c + k * c + l] = points[ii * c + l];
            }
        }
    }
}