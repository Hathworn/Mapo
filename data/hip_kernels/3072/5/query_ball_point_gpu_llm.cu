#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx, int *pts_cnt) {
    int batch_index = blockIdx.x;
    xyz1 += n * 3 * batch_index;
    xyz2 += m * 3 * batch_index;
    idx += m * nsample * batch_index;
    pts_cnt += m * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Initialize shared memory for xyz2 and idx
    __shared__ float shared_xyz2[1024 * 3]; // Assuming maximum 1024 threads per block
    __shared__ int shared_idx[1024 * nsample];

    // Load xyz2 into shared memory
    if (index < m) {
        shared_xyz2[index * 3 + 0] = xyz2[index * 3 + 0];
        shared_xyz2[index * 3 + 1] = xyz2[index * 3 + 1];
        shared_xyz2[index * 3 + 2] = xyz2[index * 3 + 2];
    }
    __syncthreads(); // Ensure all threads have loaded xyz2

    for (int j = index; j < m; j += stride) {
        float x2 = shared_xyz2[j * 3 + 0];
        float y2 = shared_xyz2[j * 3 + 1];
        float z2 = shared_xyz2[j * 3 + 2];

        float nearest_d = 1.0e99;
        int nearest_k = -1;
        int cnt = 0;

        for (int k = 0; k < n; ++k) {
            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            float d = max(sqrtf((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1)), 1e-20f);

            if (d < radius) {
                if (cnt == 0) {
                    for (int l = 0; l < nsample; ++l)
                        shared_idx[j * nsample + l] = k; // Set all indices to k
                }
                if (cnt < nsample) {
                    shared_idx[j * nsample + cnt] = k;
                    cnt++;
                }
            }
            if (d < nearest_d) {
                nearest_d = d;
                nearest_k = k;
            }
        }

        if (cnt == 0) {
            for (int l = 0; l < nsample; ++l) {
                shared_idx[j * nsample + l] = nearest_k;
            }
        }

        pts_cnt[j] = cnt;

        for (int l = 0; l < nsample; ++l) {
            idx[j * nsample + l] = shared_idx[j * nsample + l]; // Write results from shared to global memory
        }
    }
}