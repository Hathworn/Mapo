#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    outi += m * n * batch_index;
    out += m * n * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Copy from dist to out, outi in parallel
    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            out[j * n + s] = dist[j * n + s];
            outi[j * n + s] = s;
        }
    }

    __syncthreads(); // Ensure all threads complete the copy before sorting

    float *p_dist;
    for (int j = index; j < m; j += stride) {
        p_dist = out + j * n;
        // Optimized selection sort for the first k elements
        for (int s = 0; s < k; ++s) {
            int min = s;
            float min_val = p_dist[s];
            // Find the min more efficiently
            for (int t = s + 1; t < n; ++t) {
                if (p_dist[t] < min_val) {
                    min = t;
                    min_val = p_dist[t];
                }
            }
            // Swap if needed
            if (min != s) {
                p_dist[min] = p_dist[s];
                p_dist[s] = min_val;
                int tmpi = outi[j * n + min];
                outi[j * n + min] = outi[j * n + s];
                outi[j * n + s] = tmpi;
            }
        }
    }
}