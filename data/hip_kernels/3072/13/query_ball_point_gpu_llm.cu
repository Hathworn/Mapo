#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    int batch_index = blockIdx.x;
    xyz1 += n*3*batch_index;
    xyz2 += m*3*batch_index;
    idx += m*nsample*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    float radius2 = radius * radius; // Precompute radius squared

    for (int j = index; j < m; j += stride) {
        int cnt = 0;
        float x2 = xyz2[j*3+0];
        float y2 = xyz2[j*3+1];
        float z2 = xyz2[j*3+2];

        #pragma unroll  // Unroll the loop
        for (int k = 0; k < n; ++k) {
            if (cnt == nsample)
                break;  // Stop if nsample points are found
            float x1 = xyz1[k*3+0];
            float y1 = xyz1[k*3+1];
            float z1 = xyz1[k*3+2];
            float dx = x2 - x1;
            float dy = y2 - y1;
            float dz = z2 - z1;
            float d2 = dx*dx + dy*dy + dz*dz; // Calculate squared distance

            if (d2 < radius2) { // Compare squared distance
                if (cnt == 0) { // Pre-fill indices
                    for (int l = 0; l < nsample; ++l) {
                        idx[j*nsample+l] = k;
                    }
                }
                idx[j*nsample+cnt] = k;
                cnt += 1;
            }
        }
    }
}