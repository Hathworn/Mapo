#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_idx = blockIdx.x;
    int sample_idx = threadIdx.x;

    if (batch_idx < b && sample_idx < m * nsample) {
        int j = sample_idx / nsample; // Calculate m index
        int k = sample_idx % nsample; // Calculate nsample index
        int ii = idx[j * nsample + k];
        for (int l = 0; l < c; ++l) {
            // Direct indexing using calculated indices
            out[j * nsample * c + k * c + l] = points[ii * c + l];
        }
    }
}