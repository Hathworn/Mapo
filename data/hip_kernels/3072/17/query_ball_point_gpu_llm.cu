#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    // Calculate global index for each thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (index >= b) return;

    // Offset pointers based on index
    xyz1 += n * 3 * index;
    xyz2 += m * 3 * index;
    idx += m * nsample * index;

    // Iterate over each query point
    for (int j = 0; j < m; ++j) {
        int cnt = 0;
        float x2 = xyz2[j * 3 + 0];
        float y2 = xyz2[j * 3 + 1];
        float z2 = xyz2[j * 3 + 2];

        // Iterate over all points to find neighbors
        for (int k = 0; k < n; ++k) {
            if (cnt == nsample)
                break; // only pick the FIRST nsample points in the ball

            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            float dx = x2 - x1;
            float dy = y2 - y1;
            float dz = z2 - z1;
            float d = max(sqrtf(dx * dx + dy * dy + dz * dz), 1e-20f);

            if (d < radius) {
                if (cnt == 0) { 
                    // Set ALL indices to k to ensure valid indices
                    for (int l = 0; l < nsample; ++l)
                        idx[j * nsample + l] = k;
                }
                idx[j * nsample + cnt] = k;
                cnt += 1;
            }
        }
    }
}