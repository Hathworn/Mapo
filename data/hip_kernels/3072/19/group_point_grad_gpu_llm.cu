#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_index = blockIdx.x;
    int index = threadIdx.x;

    // Adjust pointers based on batch index
    idx += m * nsample * batch_index;
    grad_out += m * nsample * c * batch_index;
    grad_points += n * c * batch_index;

    // Iterate over each point in m by using thread index
    for (int j = index; j < m; j += blockDim.x) {
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            for (int l = 0; l < c; ++l) {
                // Use atomicAdd to avoid data race
                atomicAdd(&grad_points[ii * c + l], grad_out[j * nsample * c + k * c + l]);
            }
        }
    }
}