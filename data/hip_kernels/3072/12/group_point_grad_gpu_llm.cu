#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_index = blockIdx.x;  // Each block handles one batch
    int sample_index = threadIdx.x;  // Each thread handles one sample

    if (batch_index < b && sample_index < nsample) {
        const int *idx_batch = idx + batch_index * m * nsample;
        const float *grad_out_batch = grad_out + batch_index * m * nsample * c;
        float *grad_points_batch = grad_points + batch_index * n * c;
        
        for (int j = 0; j < m; ++j) {
            int ii = idx_batch[j * nsample + sample_index];
            for (int l = 0; l < c; ++l) {
                atomicAdd(&grad_points_batch[ii * c + l], grad_out_batch[j * nsample * c + sample_index * c + l]);  // Use atomicAdd for safe updates
            }
        }
    }
}