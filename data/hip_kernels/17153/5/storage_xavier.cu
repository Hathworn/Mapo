#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void storage_xavier(float *a, int size, float scale, hiprandState *cs) {
int index = blockIdx.x * blockDim.x + threadIdx.x;
if (index < size) {
hiprand_init(1234, index, 0, &cs[index]);
a[index] = (hiprand_uniform(&cs[index]) * 2 - 1) * scale;
}
}