#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv2genericrev(float *input, float *kernel, float *output, int input_n, int input_h, int input_w, int kernel_n, int kernel_h, int kernel_w, float alpha, int stride_h, int stride_w)
{
    // output dimensions
    int output_h = input_h - (kernel_h - 1) * stride_h;
    int output_w = input_w - (kernel_w - 1) * stride_w;

    // this thread only processes one output, defined by the block Ids
    int kk = blockIdx.x;
    int ii = blockIdx.y;

    // thread indices
    int batch = threadIdx.z;
    int kid = threadIdx.x;

    // compute shared memory size
    int shared_mem_size = output_h * output_w;

    // one thread only sees one output
    output = output + (kk * input_n + ii) * shared_mem_size;

    // allocate shared memory
    extern __shared__ float shared_output[];

    // generate tid outputs in shared memory
    int tid = threadIdx.x + threadIdx.z * blockDim.x;
    float *output_s = shared_output + tid * shared_mem_size;

    int yy = threadIdx.y;
    output_s += yy * output_w;
    
    // convolution loop
    for (int xx = 0; xx < output_w; xx++) {
        // Pointers for input and kernel
        float *input_p = input + (ii + batch * input_n) * input_h * input_w + yy * stride_h * input_w + xx * stride_w;
        float *kernel_p = kernel + (kk + batch * kernel_n) * kernel_w * kernel_h;
        float sum = 0;

        // Inner convolution loop using strided access
        for (int ky = 0; ky < kernel_h; ky++) {
            for (int kx = kid; kx < kernel_w; kx += blockDim.x) {
                sum += input_p[kx] * kernel_p[kx];
            }
            input_p += input_w;
            kernel_p += kernel_w;
        }
        *(output_s++) = sum;
    }
    __syncthreads();

    // reduce and write back
    if (yy == 0) {
        // reduce outputs
        for (int i = tid; i < shared_mem_size; i += blockDim.x * blockDim.z) {
            shared_output[i] = 0;
            for (int k = 0; k < blockDim.x * blockDim.z; k++) {
                shared_output[i] += shared_output[k * shared_mem_size + i];
            }
        }
        __syncthreads();

        // add existing output, and write back
        for (int i = tid; i < shared_mem_size; i += blockDim.x * blockDim.z) {
            output[i] += alpha * shared_output[i];
        }
    }
}