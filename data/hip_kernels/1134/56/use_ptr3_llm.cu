#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void use_ptr3() {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global index
    int stride = blockDim.x * gridDim.x; // Calculate stride
    for (int i = idx; i < 100; i += stride) { // Iterate with stride to cover all elements
        const_ptr[i] = i; // Assign value to global memory
    }
}