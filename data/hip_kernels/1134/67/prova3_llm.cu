#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prova3() {
    // Optimize memory access by using shared memory
    __shared__ int shared_output[32];

    // Assuming NQfrontier is a function that can be optimized further for better thread cooperativity
    auto A = NQfrontier<32>(F_array, 5, Adj_array);

    // Store results in shared memory to minimize global memory access
    for (auto it : A) {
        shared_output[threadIdx.x] = it.start;
        // Print output for debugging purposes (Comment out in release)
        // printf("threadIdx.x %d \t %d\n", threadIdx.x, it.end);
    }

    // Write results from shared memory to global output if needed
    Ouptput[threadIdx.x] = shared_output[threadIdx.x];
    // Debug statement to check the start of the first element
    // printf("threadIdx.x %d \t %d\n", threadIdx.x, (*A.begin()).start);
}