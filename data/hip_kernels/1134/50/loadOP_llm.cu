#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void loadOP() {
    // Use loop unrolling and shared memory for optimization
    __shared__ float shared_input[9];
    
    int tid = threadIdx.x;
    
    if (tid < 9) {
        shared_input[tid] = input[tid];
    }
    
    __syncthreads();
    
    if (tid == 0) output[0] = Load<DF>(shared_input[0]);
    else if (tid == 1) output[1] = Load<CA>(shared_input[1]);
    else if (tid == 2) output[2] = Load<CG>(shared_input[2]);
    else if (tid == 3) output[3] = Load<CS>(shared_input[3]);
    else if (tid == 4) output[4] = Load<CV>(shared_input[4]);
    else if (tid == 5) output[5] = Load<NC>(shared_input[5]);
    else if (tid == 6) output[6] = Load<NC_CA>(shared_input[6]);
    else if (tid == 7) output[7] = Load<NC_CG>(shared_input[7]);
    else if (tid == 8) output[8] = Load<NC_CS>(shared_input[8]);
}