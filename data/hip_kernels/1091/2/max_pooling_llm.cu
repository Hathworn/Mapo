#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void max_pooling(unsigned char* original_img, unsigned char* new_img, unsigned int width, unsigned int num_thread, unsigned int size) {
    // Calculate a unique global thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop over the elements assigned to this thread
    for (int i = tid; i < size / 4; i += num_thread) {
        unsigned int position = i + (4 * (i / 4)) + (width * 4 * (i / (width * 2)));
        // Initialize max with the first value in the pooling window
        unsigned char max = original_img[position];

        // Compare with the other values in the pooling window
        max = max > original_img[position + 4] ? max : original_img[position + 4];
        max = max > original_img[position + width] ? max : original_img[position + width];
        max = max > original_img[position + width + 4] ? max : original_img[position + width + 4];

        // Store the max value in the output image
        new_img[i] = max;
    }
}