#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rectify(unsigned char * original_img, unsigned char* new_img, unsigned int num_thread, unsigned int size) {
    // Compute the global index of the thread
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // Use stride to handle more than one element per thread if needed
    unsigned int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < size; i += stride) {
        // Apply rectification
        new_img[i] = (original_img[i] < 127) ? 127 : original_img[i];
    }
}