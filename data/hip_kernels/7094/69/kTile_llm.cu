#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Use shared memory for enhanced memory access efficiency
    extern __shared__ float sharedSrc[];

    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    // Load data into shared memory
    for (uint ty = threadIdx.x; ty < srcHeight; ty += blockDim.x) {
        for (uint tx = 0; tx < srcWidth; ++tx) {
            sharedSrc[ty * srcWidth + tx] = src[ty * srcWidth + tx];
        }
    }
    __syncthreads();

    // Process tiles
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        tgt[i] = sharedSrc[srcY * srcWidth + srcX];
    }
}