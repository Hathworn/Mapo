#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];
    
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float temp = 0.0f;

    // Load data into shared memory and calculate partial dot product
    if (eidx < numElements) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            temp += a[eidx] * b[eidx];
        }
    }

    shmem[threadIdx.x] = temp;
    __syncthreads();
    
    // Unrolling reduction using shared memory
    if (threadIdx.x < 512) { shmem[threadIdx.x] += shmem[threadIdx.x + 512]; }
    __syncthreads();
    if (threadIdx.x < 256) { shmem[threadIdx.x] += shmem[threadIdx.x + 256]; }
    __syncthreads();
    if (threadIdx.x < 128) { shmem[threadIdx.x] += shmem[threadIdx.x + 128]; }
    __syncthreads();
    if (threadIdx.x < 64) { shmem[threadIdx.x] += shmem[threadIdx.x + 64]; }
    __syncthreads();
    
    // Reduction within warp
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }
    
    // Write result to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}