#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Use boolean short-circuiting to avoid unnecessary work
        if (labelp == maxp) {
            int numMax = 1;  // Include the current label in the count
            for (int i = 0; i < numOut; i++) {
                if (i != label && probs[i * numCases + tx] == maxp) {
                    numMax++;
                }
            }
            correctProbs[tx] = 1.0f / float(numMax);
        } else {
            correctProbs[tx] = 0.0f;
        }
    }
}