#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Calculate unique thread index
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Ensure only one thread performs the division
    if (idx == 0) {
        *accuracy /= N;
    }
}