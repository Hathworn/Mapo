#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Use a block-wide stride to reduce launch overhead
    for (uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
         tidx < NUM_RND_THREADS_PER_BLOCK * gridDim.x; 
         tidx += blockDim.x * gridDim.x) {
        hiprand_init(seed, tidx, 0, &state[tidx]); // Initialize each state
    }
}