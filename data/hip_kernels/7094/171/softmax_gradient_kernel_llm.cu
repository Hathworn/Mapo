#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    
    // Reduce using warp shuffle for better performance  
    float tmp = 0;
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Use a single warp for the final reduction  
    if (idx < warpSize) {
        #pragma unroll
        for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
            if (idx < offset) {
                reduction_buffer[idx] += reduction_buffer[idx + offset];
            }
            __syncthreads();
        }
    }
    
    // Compute gradient after final reduction with the first thread managing the tmp  
    if (idx == 0) {
        tmp = reduction_buffer[0];
    }
    __syncthreads();
    
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}