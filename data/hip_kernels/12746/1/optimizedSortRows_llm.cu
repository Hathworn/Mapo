#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THRESHOLD 10010000

__device__ int cudaGetNextInColor(int *image, int x, int row, int imageWidth, int color){
    for (int i = x + 1; i < imageWidth; ++i) {
        if(THRESHOLD >= (color - image[row*imageWidth + i])){
            return i-1;
        }
    }
    return imageWidth - 1;
}

__device__ int cudaGetFirstNotInColor(int *image, int x, int row, int imageWidth, int color){
    for (int i = x; i < imageWidth; ++i) {
        if(THRESHOLD < (color - image[row*imageWidth + i])){
            return i;
        }
    }
    return -1;
}

__device__ void optimizedBubbleSort(int *pixelsToSort, int length){
    for(int i = 0; i < length; i++) {
        for(int j = 0; j < length-1; j++) {
            if(pixelsToSort[j] > pixelsToSort[j+1]){
                pixelsToSort[j] = pixelsToSort[j] ^ pixelsToSort[j+1];
                pixelsToSort[j+1] = pixelsToSort[j] ^ pixelsToSort[j+1];
                pixelsToSort[j] = pixelsToSort[j] ^ pixelsToSort[j+1];
            }
        }
    }
}

__global__ void optimizedSortRows(int *image, int imageHeight, int imageWidth, int colorMode){
    int row = blockIdx.x * blockDim.x + threadIdx.x; // Calculate row index
    if(row < imageHeight) {
        int startingX = 0;
        int finishX = 0;
        int pixelsToSort[1024]; // Use fixed-size array instead of dynamic allocation

        while(startingX < imageWidth) { // Ensure processing only relevant portions
            startingX = cudaGetFirstNotInColor(image, startingX, row, imageWidth, colorMode);
            if (startingX < 0) break; // Early exit if no more segments
            finishX = cudaGetNextInColor(image, startingX, row, imageWidth, colorMode);

            int pixelsToSortLength = min(finishX - startingX, 1024); // Simplify length calculation
            for (int i = 0; i < pixelsToSortLength; ++i) {
                pixelsToSort[i] = image[row*imageWidth + startingX + i];
            }

            optimizedBubbleSort(pixelsToSort, pixelsToSortLength);

            for (int i = 0; i < pixelsToSortLength; ++i) {
                image[row*imageWidth + startingX + i] = pixelsToSort[i];
            }

            startingX = finishX + 1; // Move to next segment
        }
    }
}