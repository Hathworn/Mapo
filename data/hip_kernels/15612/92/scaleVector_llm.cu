#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Calculate global thread index
    int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a while loop to handle stragglers in a single thread
    while (pos < len) {
        d_res[pos] = d_src[pos] * scale;
        pos += gridDim.x * blockDim.x; // Move to next stride
    }
}