#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure valid index and use register for scale multiplication
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];
        float locX = loc.x * scale;
        float locY = loc.y * scale;

        // Store results in global memory
        x[ptidx] = locX;
        y[ptidx] = locY;
    }
}