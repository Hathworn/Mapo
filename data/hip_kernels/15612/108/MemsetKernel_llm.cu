#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global thread index for 1D flattened array
    int idx = blockIdx.y * blockDim.y * w + blockIdx.x * blockDim.x + threadIdx.y * w + threadIdx.x;

    // Grid stride loop for handling images larger than grid
    const int stride = blockDim.x * gridDim.x; 
    for (int pos = idx; pos < w * h; pos += stride) {
        image[pos] = value; 
    }
}