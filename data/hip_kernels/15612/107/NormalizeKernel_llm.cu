#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread does not work on out-of-bounds data
    if (i < h && j < w)
    {
        const int pos = i * s + j;

        float scale = normalization_factor[pos];

        // Use ternary operation to avoid division by zero
        float invScale = (scale != 0.0f) ? (1.0f / scale) : 1.0f;

        // Directly update the image value with the calculated inverse scale
        image[pos] *= invScale;
    }
}