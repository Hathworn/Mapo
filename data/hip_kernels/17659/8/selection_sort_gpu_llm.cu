#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    outi += m * n * batch_index;
    out += m * n * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Copy from dist to dist_out
    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            out[j * n + s] = dist[j * n + s];
            outi[j * n + s] = s;
        }
    }

    // Optimize the selection sort loop
    __shared__ float shared_dist[1024]; // Use shared memory to accelerate access
    __shared__ int shared_outi[1024];

    for (int j = index; j < m; j += stride) {
        if (threadIdx.x < n) {
            shared_dist[threadIdx.x] = out[j * n + threadIdx.x];
            shared_outi[threadIdx.x] = outi[j * n + threadIdx.x];
        }
        __syncthreads();

        // Improved selection sort for the first k elements
        for (int s = 0; s < k; ++s) {
            int min = s;
            for (int t = s + 1; t < n; ++t) {
                if (shared_dist[t] < shared_dist[min]) {
                    min = t;
                }
            }
            if (min != s) {
                float temp = shared_dist[min];
                shared_dist[min] = shared_dist[s];
                shared_dist[s] = temp;
                int temp_i = shared_outi[min];
                shared_outi[min] = shared_outi[s];
                shared_outi[s] = temp_i;
            }
        }

        if (threadIdx.x < n) {
            out[j * n + threadIdx.x] = shared_dist[threadIdx.x];
            outi[j * n + threadIdx.x] = shared_outi[threadIdx.x];
        }
        __syncthreads();
    }
}