#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, float *dist, int *idx, float *val) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    idx += m * k * batch_index;
    val += m * k * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Pre-fetch data to shared memory for faster access
    extern __shared__ float shared_dist[];
    for (int j = index; j < m; j += stride) {
        float *p_dist = dist + j * n;

        // Load current row into shared memory
        for (int t = index; t < n; t += stride) {
            shared_dist[j * n + t] = p_dist[t];
        }
        __syncthreads();

        // Perform selection sort on shared memory for the first k elements
        for (int s = 0; s < k; ++s) {
            int min = s;
            for (int t = s + 1; t < n; ++t) {
                if (shared_dist[j * n + t] < shared_dist[j * n + min]) {
                    min = t;
                }
            }
            // Write results back to global memory
            idx[j * n + s] = min;
            val[j * n + s] = shared_dist[j * n + min];

            // Swap min-th and s-th element in shared memory
            float tmp = shared_dist[j * n + min];
            shared_dist[j * n + min] = shared_dist[j * n + s];
            shared_dist[j * n + s] = tmp;
        }
        __syncthreads(); // Ensure all threads have committed changes
    }
}