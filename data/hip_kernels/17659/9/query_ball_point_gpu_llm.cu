#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    // Use blockIdx.x for batch index (i) and blockIdx.y for the point index (j) in xyz2
    int i = blockIdx.x;
    int j = blockIdx.y;
    
    if (i < b && j < m) {
        int cnt = 0;
        float x2 = xyz2[j * 3 + 0];
        float y2 = xyz2[j * 3 + 1];
        float z2 = xyz2[j * 3 + 2];

        for (int k = 0; k < n; ++k) {
            if (cnt == nsample) break; // Stop when nsample points are collected

            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            // Compute squared distance to avoid sqrt computation
            float d2 = (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1);
            if (d2 < radius * radius) {
                if (cnt == 0) { // Initialize all indices to k
                    for (int l = 0; l < nsample; ++l) {
                        idx[j * nsample + l] = k;
                    }
                }
                idx[j * nsample + cnt] = k;
                cnt += 1;
            }
        }
    }
}