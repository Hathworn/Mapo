#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Use global index for parallel execution
    if (index >= b) return; // Exit if index exceeds bounds

    xyz1 += n * 3 * index;
    xyz2 += m * 3 * index;
    idx += m * nsample * index;

    for (int j = 0; j < m; ++j) {
        int cnt = 0;
        float x2 = xyz2[j * 3 + 0];
        float y2 = xyz2[j * 3 + 1];
        float z2 = xyz2[j * 3 + 2];

        for (int k = 0; k < n; ++k) {
            if (cnt == nsample) break; // Break if nsample points are found

            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            float dx = x2 - x1;
            float dy = y2 - y1;
            float dz = z2 - z1; // Calculate differences
            float d = sqrtf(dx * dx + dy * dy + dz * dz);

            if (d < radius) {
                if (cnt == 0) {
                    for (int l = 0; l < nsample; ++l) {
                        idx[j * nsample + l] = k; // Initialize all indices
                    }
                }
                idx[j * nsample + cnt] = k; // Assign index
                cnt += 1;
            }
        }
    }
}