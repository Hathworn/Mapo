#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    // Calculate global thread indices
    int batch_index = blockIdx.z;
    int m_index = blockIdx.y * blockDim.y + threadIdx.y;
    int sample_index = blockIdx.x * blockDim.x + threadIdx.x;

    if (batch_index < b && m_index < m && sample_index < nsample) {
        int ii = idx[m_index * nsample + sample_index];
        for (int l = 0; l < c; ++l) {
            out[(m_index * nsample + sample_index) * c + l] = points[ii * c + l];
        }
    }
}