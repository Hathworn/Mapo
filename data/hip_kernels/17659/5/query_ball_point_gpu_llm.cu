#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx, int *pts_cnt) {
    int batch_index = blockIdx.x;
    xyz1 += n * 3 * batch_index;
    xyz2 += m * 3 * batch_index;
    idx += m * nsample * batch_index;
    pts_cnt += m * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int j = index; j < m; j += stride) {
        int cnt = 0;
        float x2 = xyz2[j * 3 + 0]; // Move these variable definitions out of the loop for optimization
        float y2 = xyz2[j * 3 + 1];
        float z2 = xyz2[j * 3 + 2];

        for (int k = 0; k < n && cnt < nsample; ++k) { // Combine loop condition with break
            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];

            float distX = x2 - x1;
            float distY = y2 - y1;
            float distZ = z2 - z1;
            float d = sqrtf(distX * distX + distY * distY + distZ * distZ);

            if (d < radius) {
                if (cnt == 0) {
                    for (int l = 0; l < nsample; ++l) {
                        idx[j * nsample + l] = k;
                    }
                }
                idx[j * nsample + cnt] = k;
                cnt++;
            }
        }
        pts_cnt[j] = cnt;
    }
}