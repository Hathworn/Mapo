#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    // Calculate the global thread index
    int batch_index = blockIdx.x;
    int thread_index = threadIdx.x;
    int index = batch_index * blockDim.x + thread_index;

    if (index >= b) return;  // Early exit for out-of-bounds threads

    // Adjust pointers for this batch
    points += n * c * index;
    idx += m * nsample * index;
    out += m * nsample * c * index;

    for (int j = 0; j < m; ++j) {
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            for (int l = 0; l < c; ++l) {
                out[j * nsample * c + k * c + l] = points[ii * c + l];  // Copy points to output
            }
        }
    }
}