#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    // Calculate the batch offset for current block
    int batch_index = blockIdx.x;
    points += n*c*batch_index;
    idx += m*nsample*batch_index;
    out += m*nsample*c*batch_index;

    // Calculate the global thread ID
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;

    // Loop across all samples using the global thread ID for coalesced memory access
    for (int j = index; j < m * nsample; j += total_threads) {
        int sample_index = j / nsample;
        int nsample_index = j % nsample;
        int ii = idx[sample_index * nsample + nsample_index];
        
        // Directly copy the vector of `c` dimension
        for (int l = 0; l < c; ++l) {
            out[sample_index * nsample * c + nsample_index * c + l] = points[ii * c + l];
        }
    }
}