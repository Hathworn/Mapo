#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    // Calculate global thread index directly
    int batch_index = blockIdx.x;
    idx += m * nsample * batch_index;
    grad_out += m * nsample * c * batch_index;
    grad_points += n * c * batch_index;

    // Pre-compute threads and limits
    int global_thread_index = threadIdx.x + blockDim.x * blockIdx.x;
    int total_threads = gridDim.x * blockDim.x;

    // Traverse elements using global index stride pattern
    for (int j = global_thread_index; j < m; j += total_threads) {
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            for (int l = 0; l < c; ++l) {
                // Utilize atomicAdd for safe updates to shared memory
                atomicAdd(&grad_points[ii * c + l], grad_out[j * nsample * c + k * c + l]);
            }
        }
    }
}