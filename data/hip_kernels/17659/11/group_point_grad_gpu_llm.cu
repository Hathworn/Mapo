#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    // Calculate global thread ID
    int batchIdx = blockIdx.x;
    int idxStride = m * nsample;
    int gradOutStride = idxStride * c;
    int gradPointsStride = n * c;

    if (batchIdx < b) {
        grad_out += batchIdx * gradOutStride;
        idx += batchIdx * idxStride;
        grad_points += batchIdx * gradPointsStride;

        // Use shared memory to reduce memory accesses
        extern __shared__ int sharedIdx[];
        for (int j = threadIdx.x; j < idxStride; j += blockDim.x) {
            sharedIdx[j] = idx[j];
        }
        __syncthreads();

        // Optimize loop ordering and memory access
        for (int j = threadIdx.x; j < m; j += blockDim.x) {
            for (int k = 0; k < nsample; ++k) {
                int ii = sharedIdx[j * nsample + k];

                for (int l = 0; l < c; ++l) {
                    atomicAdd(&grad_points[ii * c + l], grad_out[j * nsample * c + k * c + l]);
                }
            }
        }
    }
}