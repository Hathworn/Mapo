#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    int batch_index = blockIdx.x;
    xyz1 += n*3*batch_index;
    xyz2 += m*3*batch_index;
    idx += m*nsample*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Cache radius square to avoid recomputation and use square distance to avoid sqrt
    float radius_square = radius * radius;

    for (int j=index; j<m; j+=stride) {
        int cnt = 0;
        float x2 = xyz2[j*3+0];
        float y2 = xyz2[j*3+1];
        float z2 = xyz2[j*3+2];

        for (int k=0; k<n; ++k) {
            float x1 = xyz1[k*3+0];
            float y1 = xyz1[k*3+1];
            float z1 = xyz1[k*3+2];

            // Compute squared distance
            float dist_square = (x2-x1)*(x2-x1) + (y2-y1)*(y2-y1) + (z2-z1)*(z2-z1);

            if (dist_square < radius_square) {
                if (cnt == 0) {
                    for (int l=0; l<nsample; ++l)
                        idx[j*nsample+l] = k;
                }
                if (cnt < nsample) {
                    idx[j*nsample+cnt] = k;
                    ++cnt;
                }
                else {
                    break; // early exit if nsample points are found
                }
            }
        }
    }
}