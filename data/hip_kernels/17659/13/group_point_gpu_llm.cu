#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n * c * batch_index;
    idx += m * nsample * batch_index;
    out += m * nsample * c * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int j = index; j < m; j += stride) {
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            const float* point_ptr = &points[ii * c]; // Precompute point address
            float* out_ptr = &out[j * nsample * c + k * c]; // Precompute output address

            for (int l = 0; l < c; ++l) {
                out_ptr[l] = point_ptr[l]; // Use precomputed addresses
            }
        }
    }
}