#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    outi += m * n * batch_index;
    out += m * n * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Efficient copying using shared memory
    extern __shared__ float shared_dist[];
    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            out[j * n + s] = dist[j * n + s];
            outi[j * n + s] = s;
            shared_dist[j * n + s] = out[j * n + s];
        }
    }
    __syncthreads();

    float *p_dist;
    for (int j = index; j < m; j += stride) {
        p_dist = shared_dist + j * n;
        // Optimize selection sort by reducing if-statement overhead
        for (int s = 0; s < k; ++s) {
            int min = s;
            float min_val = p_dist[s];
            // Unroll loop for small performance gain
            #pragma unroll
            for (int t = s + 1; t < n; ++t) {
                if (p_dist[t] < min_val) {
                    min = t;
                    min_val = p_dist[t];
                }
            }
            // Avoid swap if unnecessary
            if (min != s) {
                float tmp = p_dist[min];
                p_dist[min] = p_dist[s];
                p_dist[s] = tmp;
                int tmpi = outi[j * n + min];
                outi[j * n + min] = outi[j * n + s];
                outi[j * n + s] = tmpi;
            }
        }
    }
}