#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_idx = blockIdx.x;
    int thread_idx = threadIdx.x;

    // Adjust pointers for batch handling
    idx += m * nsample * batch_idx;
    grad_out += m * nsample * c * batch_idx;
    grad_points += n * c * batch_idx;

    // Parallelize over 'm' using blockDim.y
    for (int j = thread_idx; j < m; j += blockDim.y) {
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            for (int l = 0; l < c; ++l) {
                atomicAdd(&grad_points[ii * c + l], grad_out[j * nsample * c + k * c + l]);
            }
        }
    }
}