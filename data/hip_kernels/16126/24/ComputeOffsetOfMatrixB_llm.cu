#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Calculate global index
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure the index is within bounds
    if (i < N) {
        *(output + blockIdx.x * N + i) = -row_sum[blockIdx.x];
    }
}