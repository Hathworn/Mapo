#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global function
__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Cache batch_index and class_index in registers for faster access
    int64_t batch_idx = batch_index;
    int64_t class_idx = class_index;
    
    // Calculate stride and iterate over elements in stride manner to balance load
    int stride = blockDim.x * gridDim.x;
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_elements; idx += stride) {
        int base_idx = idx * 3;
        
        // Perform memory coalesced writes
        to_normalize[base_idx] = batch_idx;
        to_normalize[base_idx + 1] = class_idx;
        to_normalize[base_idx + 2] = static_cast<int64_t>(original[idx]);
    }
}