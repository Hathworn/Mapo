#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    int32_t i = threadIdx.x + blockIdx.x * blockDim.x;
    // Launch as many threads as elements to ensure all elements are processed
    if (i < N) {
        output[blockIdx.x * N + i] = -col_sum[i];
    }
}