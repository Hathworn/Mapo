#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
    // Cache the row sum for the current block to reduce global memory access
    int32_t row_sum_value = row_sum[blockIdx.x];
    
    // Use grid-stride loop to handle larger datasets
    for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        output[blockIdx.x * N + i] = K_A_B - row_sum_value - col_sum[i];
    }
}