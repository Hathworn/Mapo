#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float compute(int idx, float* buf, int s)
{
    // Some random calcs to make the kernel unempty
    float k = 0.0f;
    for (int x = 0; x < s; x++) {
        k += cosf(x * 0.1f * idx);
        buf[x] = k;
    }
    
    // Optimize: Combine loops to improve spatial locality
    float sum = 0.0f;
    for (int x = s - 1; x >= 0; x--) {
        buf[x] = buf[x] * buf[x];
        if (x > 0) {
            sum += buf[x - 1] / (fabsf(buf[x]) + 0.1f);
        }
    }
    
    return sum;
}

__global__ void testWithGlobal(int n, int s, float* result, float* buf) {
    // Calculate unique thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        // Optimize: Use shared memory for buffer per block
        extern __shared__ float sharedBuf[];
        float* localBuf = &sharedBuf[threadIdx.x * s];
        
        result[idx] = compute(idx, localBuf, s);
    }
}