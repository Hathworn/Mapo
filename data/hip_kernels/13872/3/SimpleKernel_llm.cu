#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SimpleKernel(int N, float* a){
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a register for the repeated value
    if (idx < N) {
        float temp = a[idx]; // Cache the initial value
        for (int x = 0; x < 1000; x++) {
            temp = asin(temp + x); // Use the register instead of global memory
        }
        a[idx] = temp; // Write back to global memory only once
    }
}