#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*!
* Copyright (c) 2017 Microsoft
* Licensed under The MIT License [see LICENSE for details]
* \file deformable_psroi_pooling.cu
* \brief
* \author Yi Li, Guodong Zhang, Jifeng Dai
*/
/***************** Adapted by Charles Shang *********************/

#define CUDA_KERNEL_LOOP(i, n)                        \
for (int i = blockIdx.x * blockDim.x + threadIdx.x; \
i < (n);                                       \
i += blockDim.x * gridDim.x)

const int CUDA_NUM_THREADS = 1024;
__device__ float bilinear_interp( const float *data, const float x, const float y, const int width, const int height)
{
int x1 = floor(x);
int x2 = ceil(x);
int y1 = floor(y);
int y2 = ceil(y);
float dist_x = (float)(x - x1);
float dist_y = (float)(y - y1);
float value11 = data[y1 * width + x1];
float value12 = data[y2 * width + x1];
float value21 = data[y1 * width + x2];
float value22 = data[y2 * width + x2];
float value = (1 - dist_x) * (1 - dist_y) * value11 + (1 - dist_x) * dist_y * value12 + dist_x * (1 - dist_y) * value21 + dist_x * dist_y * value22;
return value;
}
__device__ double bilinear_interp( const double *data, const double x, const double y, const int width, const int height)
{
int x1 = floor(x);
int x2 = ceil(x);
int y1 = floor(y);
int y2 = ceil(y);
double dist_x = (double)(x - x1);
double dist_y = (double)(y - y1);
double value11 = data[y1 * width + x1];
double value12 = data[y2 * width + x1];
double value21 = data[y1 * width + x2];
double value22 = data[y2 * width + x2];
double value = (1 - dist_x) * (1 - dist_y) * value11 + (1 - dist_x) * dist_y * value12 + dist_x * (1 - dist_y) * value21 + dist_x * dist_y * value22;
return value;
}

__global__ void DeformablePSROIPoolForwardKernel( const int count, const double *bottom_data, 
    const double spatial_scale, const int channels, const int height, const int width, 
    const int pooled_height, const int pooled_width, const double *bottom_rois, 
    const double *bottom_trans, const int no_trans, const double trans_std, 
    const int sample_per_part, const int output_dim, const int group_size, 
    const int part_size, const int num_classes, const int channels_each_class,
    double *top_data, double *top_count)
{
    CUDA_KERNEL_LOOP(index, count)
    {
        // Calculate position indices using modulo to improve access pattern
        int pw = index % pooled_width;
        int ph = (index / pooled_width) % pooled_height;
        int ctop = (index / pooled_width / pooled_height) % output_dim;
        int n = index / pooled_width / pooled_height / output_dim;

        // Access bottom_rois with offset
        const double *offset_bottom_rois = bottom_rois + n * 5;
        int roi_batch_ind = offset_bottom_rois[0];
        double roi_start_w = round(offset_bottom_rois[1]) * spatial_scale - 0.5;
        double roi_start_h = round(offset_bottom_rois[2]) * spatial_scale - 0.5;
        double roi_end_w = (round(offset_bottom_rois[3]) + 1.) * spatial_scale - 0.5;
        double roi_end_h = (round(offset_bottom_rois[4]) + 1.) * spatial_scale - 0.5;

        // Force small ROIs to be 1x1 and calculate bin sizes
        double roi_width = max(roi_end_w - roi_start_w, 0.1);
        double roi_height = max(roi_end_h - roi_start_h, 0.1);
        double bin_size_h = roi_height / pooled_height;
        double bin_size_w = roi_width / pooled_width;
        double sub_bin_size_h = bin_size_h / sample_per_part;
        double sub_bin_size_w = bin_size_w / sample_per_part;

        // Calculate transformed positions
        int part_h = floor(ph / (double)pooled_height * part_size);
        int part_w = floor(pw / (double)pooled_width * part_size);
        int class_id = ctop / channels_each_class;
        double trans_x = no_trans ? 0 : bottom_trans[(((n * num_classes + class_id) * 2) * part_size + part_h) * part_size + part_w] * trans_std;
        double trans_y = no_trans ? 0 : bottom_trans[(((n * num_classes + class_id) * 2 + 1) * part_size + part_h) * part_size + part_w] * trans_std;
        
        double wstart = pw * bin_size_w + roi_start_w + trans_x * roi_width;
        double hstart = ph * bin_size_h + roi_start_h + trans_y * roi_height;

        double sum = 0;
        int count = 0;
        int gw = min(max(floor(pw * (double)group_size / pooled_width), 0), group_size - 1);
        int gh = min(max(floor(ph * (double)group_size / pooled_height), 0), group_size - 1);

        const double *offset_bottom_data = bottom_data + roi_batch_ind * channels * height * width;

        // Optimize nested loops by making iterations explicit
        for (int ih = 0; ih < sample_per_part; ih++) 
        {
            for (int iw = 0; iw < sample_per_part; iw++)
            {
                double w = wstart + iw * sub_bin_size_w;
                double h = hstart + ih * sub_bin_size_h;
                if (w >= -0.5 && w <= width - 0.5 && h >= -0.5 && h <= height - 0.5)
                {
                    w = min(max(w, 0.), width - 1.);
                    h = min(max(h, 0.), height - 1.);
                    int c = (ctop * group_size + gh) * group_size + gw;
                    double val = bilinear_interp(offset_bottom_data + c * height * width, w, h, width, height);
                    sum += val;
                    count++;
                }
            }
        }

        top_data[index] = count == 0 ? 0 : sum / count;
        top_count[index] = count;
    }
}