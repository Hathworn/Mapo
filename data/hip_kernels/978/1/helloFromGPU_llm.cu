#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to print from a specific thread
__global__ void helloFromGPU(void)
{
    // Use a single if statement with a return to optimize performance
    if (threadIdx.x != 5) return;
    printf("Hello World from GPU thread %d!\n", threadIdx.x);
}