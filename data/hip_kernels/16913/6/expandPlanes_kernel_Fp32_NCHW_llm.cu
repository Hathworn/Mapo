#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expandPlanes_kernel_Fp32_NCHW(float* output, const uint64_t* masks, const float* values, int n) {
    // Block size of 256, same mask/val for 64 consecutive threads.
    constexpr int kNumShmemElements = 256 / 64;

    __shared__ uint64_t shMasks[kNumShmemElements];
    __shared__ float shVals[kNumShmemElements];

    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int planeIndex = index >> 6;
    int shmemIdx = threadIdx.x >> 6;  // Calculated once

    if (planeIndex >= n) return;

    // Load inputs to shared memory once per 64 threads.
    if (threadIdx.x < kNumShmemElements) {
        shMasks[threadIdx.x] = masks[planeIndex + threadIdx.x];
        shVals[threadIdx.x] = values[planeIndex + threadIdx.x];
    }
    __syncthreads();

    uint64_t mask = shMasks[shmemIdx];

    int sqIndex = index & 0x3F;
    float op = 0;

    // Use the more concise variable and calculation.
    if (mask & (1ull << sqIndex)) {
        op = shVals[shmemIdx];
    }
    output[index] = op;
}