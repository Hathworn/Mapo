#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc(float *d_D, int n, int k) {
    __shared__ float s_d[3*256]; // Shared memory allocation

    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global i index
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Calculate global j index

    int b_index = 3 * (threadIdx.x + blockDim.x*threadIdx.y); // Calculate index in shared memory
    if (i < n && j < n) { // Ensure i and j are within bounds
        s_d[b_index] = d_D[i + j*n]; // Load data into shared memory
        s_d[b_index + 1] = d_D[i + k*n]; // Load data into shared memory
        s_d[b_index + 2] = d_D[k + j*n]; // Load data into shared memory

        __syncthreads(); // Synchronize threads within block

        if (s_d[b_index] > s_d[b_index + 1] + s_d[b_index + 2]) {
            s_d[b_index] = s_d[b_index + 1] + s_d[b_index + 2]; // Compute minimum distance
        }

        __syncthreads(); // Synchronize before writing to global memory

        d_D[i + j*n] = s_d[b_index]; // Write back to global memory
    }
}