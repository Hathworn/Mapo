#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef INFINITY
/* INFINITY is supported */
#endif

float **A, **D, *d2; // Table A distance, D minimum distance, d2 tempTable 1-d

__global__ void calc(float *d_D, int n, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure i and j are within bounds
    if (i < n && j < n) {
        float new_dist = d_D[i + k * n] + d_D[k + j * n];
        // Use local variable to minimize global memory access
        float current_dist = d_D[i + j * n];
        if (current_dist > new_dist) {
            d_D[i + j * n] = new_dist;
        }
    }
}