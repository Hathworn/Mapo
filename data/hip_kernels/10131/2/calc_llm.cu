#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc(float *d_D, int n, int k) {
    __shared__ float s_d[4*3*256]; // Shared table within a block
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculation of i and j
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int b_index = 4 * 3 * (threadIdx.x + blockDim.x * threadIdx.y); // Shared mem index
    int istep = blockDim.x * gridDim.x, jstep = blockDim.y * gridDim.y;
    
    int l, m, v = 0;
    
    // Loop unrolling for loading data into shared memory
    #pragma unroll
    for (l = 0; l < 2; l++) {
        #pragma unroll
        for (m = 0; m < 2; m++) {
            s_d[b_index + 3 * v] = d_D[(i + l * istep) + (j + m * jstep) * n];
            s_d[b_index + (3 * v + 1)] = d_D[(i + l * istep) + k * n];
            s_d[b_index + (3 * v + 2)] = d_D[k + (j + m * jstep) * n];
            v++;
        }
    }
    
    // Compute new cell values with branchless conditional logic
    #pragma unroll
    for (v = 0; v < 4; v++) {
        float sum = s_d[b_index + (3 * v + 1)] + s_d[b_index + (3 * v + 2)];
        float original = s_d[b_index + 3 * v];
        s_d[b_index + 3 * v] = fminf(original, sum); // Use fminf intrinsic for branchless min
    }
    
    v = 0;
    // Loop unrolling for storing data back to global memory
    #pragma unroll
    for (l = 0; l < 2; l++) {
        #pragma unroll
        for (m = 0; m < 2; m++) {
            d_D[(i + l * istep) + (j + m * jstep) * n] = s_d[b_index + 3 * v];
            v++;
        }
    }
}