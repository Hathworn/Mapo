#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = __float2int_rd(labels[tx]); // Use faster intrinsic for float to int conversion
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Optimized conditional statement
        if (labelp == maxp) {
            int numMax = 0;

            // Unroll the loop for potential performance improvement
            for (int i = 0; i < numOut; ++i) {
                numMax += __fastequf(probs[i * numCases + tx], maxp); // Use fast float equality check
            }
            correctProbs[tx] = 1.0f / float(numMax);
        } else {
            correctProbs[tx] = 0.0f;
        }
    }
}