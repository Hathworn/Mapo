#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread 0 to avoid race condition, only a single division needed
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}