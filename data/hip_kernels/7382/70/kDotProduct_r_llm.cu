#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0; // Use a register to accumulate results

    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        sum += a[eidx] * b[eidx];
    }
    shmem[threadIdx.x] = sum; // Save register sum to shared memory
    __syncthreads();

    // Optimize reduction using unrolling
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();

    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();

    if (threadIdx.x < 32) {
        // Use warp shuffle for final reduction
        float val = shmem[threadIdx.x];
        val += __shfl_down(val, 16);
        val += __shfl_down(val, 8);
        val += __shfl_down(val, 4);
        val += __shfl_down(val, 2);
        val += __shfl_down(val, 1);
        if (threadIdx.x == 0) {
            target[blockIdx.x] = val;
        }
    }
}