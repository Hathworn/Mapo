#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    // Efficient parallel initialization using shared memory to reduce global memory access
    __shared__ hiprandState localState[NUM_RND_THREADS_PER_BLOCK];
    
    // Initialize state in shared memory
    hiprand_init(seed, tidx, 0, &localState[threadIdx.x]);
    
    // Copy state from shared to global memory
    state[tidx] = localState[threadIdx.x];
}