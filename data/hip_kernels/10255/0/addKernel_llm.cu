#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//-----include header files, ¤Þ¤J¼ÐÀYÀÉ-----

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

// Optimize kernel for better data access and parallelism
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global index 
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (i < N) { 
        c[i] = a[i] + b[i];
    }
}