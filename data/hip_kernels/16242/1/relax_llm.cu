#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relax(int* U, int* F, int* d, size_t gSize, int* adjMat) {
    // Calculate global thread ID
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread ID is valid and the node is fixed
    if (globalThreadId < gSize && F[globalThreadId]) {
        for (int i = 0; i < gSize; i++) {
            // Check adjacency and ensure not self-loop, node should be unrelaxed
            if (adjMat[globalThreadId * gSize + i] && i != globalThreadId && U[i]) {
                // Use atomicMin to minimize the distance
                atomicMin(&d[i], d[globalThreadId] + adjMat[globalThreadId * gSize + i]);
            }
        }
    }
}