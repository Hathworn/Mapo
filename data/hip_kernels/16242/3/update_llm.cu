#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update(int* U, int* F, int* d, int* del, size_t gSize) {
    // Calculate the global thread ID
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread is within bounds
    if (globalThreadId < gSize) {
        int u_val = U[globalThreadId];
        int d_val = d[globalThreadId];
        
        // Use a single conditional assignment with ternary operator
        F[globalThreadId] = (u_val && d_val < del[0]) ? 1 : 0;

        // Update U only when condition is met
        if (F[globalThreadId]) {
            U[globalThreadId] = 0;
        }
    }
}