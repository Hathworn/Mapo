#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global memory access and avoid branching
__global__ void init(int* __restrict__ U, int* __restrict__ F, int* __restrict__ d, int startNode, size_t gSize) {
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

    if (globalThreadId < gSize) {
        // Use a single initialization statement
        U[globalThreadId] = (globalThreadId == 0) ? 0 : 1;
        F[globalThreadId] = (globalThreadId == 0) ? 1 : 0;
        d[globalThreadId] = (globalThreadId == 0) ? 0 : INT_MAX;
    }
}