#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findAllMins(int* adjMat, int* outVec, size_t gSize) {
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    if(globalThreadId < gSize) {
        int ind = globalThreadId * gSize;
        int min = INT_MAX;

        // Use shared memory to reduce global memory accesses
        __shared__ int sharedAdjMat[1024]; // Assuming blockDim.x <= 1024

        for(int i = threadIdx.x; i < gSize; i += blockDim.x) {
            sharedAdjMat[i] = adjMat[ind + i];
        }
        __syncthreads();

        // Each thread finds the local minimum
        for(int i = 0; i < gSize; i++) {
            if(sharedAdjMat[i] < min && sharedAdjMat[i] > 0) {
                min = sharedAdjMat[i];
            }
        }

        outVec[globalThreadId] = min;
    }
}