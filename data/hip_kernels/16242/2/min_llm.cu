#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void min(int* U, int* d, int* outDel, int* minOutEdges, size_t gSize, int useD) {
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate positions in the input array for the current thread
    int pos1 = 2 * globalThreadId;
    int pos2 = pos1 + 1;

    // Temporary variables to store values and ensure single branch computation for outDel
    int val1 = INT_MAX, val2 = INT_MAX;

    // Process pos1 if within bounds
    if (pos1 < gSize) {
        val1 =  minOutEdges[pos1] + (useD ? d[pos1] : 0);
        val1 = val1 <= 0 ? INT_MAX : val1;
        if (useD) {
            val1 = U[pos1] ? val1 : INT_MAX;
        }
    }

    // Process pos2 if within bounds
    if (pos2 < gSize) {
        val2 = minOutEdges[pos2] + (useD ? d[pos2] : 0);
        val2 = val2 <= 0 ? INT_MAX : val2;
        if (useD) {
            val2 = U[pos2] ? val2 : INT_MAX;
        }
    }

    // Write the minimum value of the two positions to output array
    outDel[globalThreadId] = min(val1, val2);
}