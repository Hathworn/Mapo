#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 100000000

__global__ void vec_sum(float* a, float* b, float* c) {
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is within bounds
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}