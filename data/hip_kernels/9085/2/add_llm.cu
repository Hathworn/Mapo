#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c) { 
    // Use shared memory for faster access
    extern __shared__ int shared_mem[];
    
    // Compute the sum and store it in shared memory
    shared_mem[0] = a + b;

    // Write the result from shared memory to global memory
    *c = shared_mem[0];
}