#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

int Nx, Ny, npts;
int *active;

__global__ void laplace_2d_gpu(double *w, double *v, const int nx, const int ny)
{
    // Calculate global indices
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Ensure processing only inner points
    if (ix > 0 && ix < (nx + 1) && iy > 0 && iy < (ny + 1)) 
    {
        // Optimize index calculation by reducing multiplications
        unsigned int idx = iy * (nx + 2) + ix;
        
        // Laplace calculation
        w[idx] = 4.0 * v[idx] - (v[idx - 1] + v[idx + 1] + v[idx - (nx + 2)] + v[idx + (nx + 2)]);
    }
}