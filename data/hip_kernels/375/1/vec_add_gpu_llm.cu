#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Global variables available in all functions */
int Nx, Ny, npts;
int *active;

/* Optimize the kernel function */
__global__ void vec_add_gpu(double *sum, double *w, double a, double *v, const int nx, const int ny)
{
    // Calculate global index
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Check if within bounds and calculate index only if active
    if (ix > 0 && ix < (nx + 1) && iy > 0 && iy < (ny + 1))
    {
        unsigned int idx = iy * (nx + 2) + ix; // Optimize index calculation
        sum[idx] = w[idx] + a * v[idx]; // Perform addition
    }
}