#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use blockIdx.x and threadIdx.x to ensure only one thread updates
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}