#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiply_device(double *d_a, double *d_b, int dim) {
    // Retrieve the thread and block specific information.
    int i = threadIdx.x;
    int j_start = blockIdx.x;
    int j_stride = gridDim.x;

    // Begin Matrix Computation.
    for (int j = j_start; j < dim; j += j_stride) {
        double sum = 0.0; // Initialize sum inside the loop for each column
        for (int k = 0; k < dim; k++) {
            double a = d_a[k * dim + i]; // Load element from d_a
            double b = d_a[k * dim + j]; // Load corresponding element from d_a
            sum += a * b; // Accumulate product
        }
        d_b[i * dim + j] = sum; // Store computed sum in the result matrix
    }
}