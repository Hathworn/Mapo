#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiply_device (double *d_a, double *d_b, int dim) {

    // Declaration of required variables.
    double sum;

    // Retrieve the thread and block specific information.
    int i = threadIdx.x;

    // Begin Matrix Computation.
    for (int j = blockIdx.x; j < dim; j += gridDim.x) {
        sum = 0;
        for (int k = 0; k < dim; k++) {
            // Precompute index to avoid repeated calculations.
            int indexOffset = k * dim;
            double a = d_a[indexOffset + i];
            double b = d_a[indexOffset + j];
            sum += a * b;
        }
        // Store the result in d_b.
        d_b[i * dim + j] = sum;
    }
}