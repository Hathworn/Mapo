#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multiply_device (double *d_a, double *d_b, int dim) {
    
    // Utilize shared memory to improve data reuse
    __shared__ double shared_a[1024];  // Assuming maximum block size
    __shared__ double shared_b[1024];
    
    // Retrieve the thread and block specific information
    int i = threadIdx.x;
    int j, k;
    
    // Start Matrix Computation by processing row-wise with shared memory
    for (j = blockIdx.x * blockDim.x + threadIdx.x; j < dim; j += blockDim.x * gridDim.x) {
        double sum = 0.0;
        for (k = 0; k < dim; k++) {
            // Load data into shared memory
            shared_a[threadIdx.x] = d_a[k * dim + i];
            shared_b[threadIdx.x] = d_a[k * dim + j];
            __syncthreads();  // Ensure all threads have loaded the data

            // Reuse shared data if all multiple of blockDim.x
            sum += shared_a[threadIdx.x] * shared_b[threadIdx.x];
            __syncthreads();  // Ensure all threads have completed computation
        }
        d_b[i * dim + j] = sum;
    }
}