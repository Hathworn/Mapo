#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void horspool_match(char* text, char* pattern, int* shift_table, unsigned int* num_matches, int chunk_size, int num_chunks, int text_size, int pat_len, unsigned int* d_output) {
    extern __shared__ int s[];

    int count = 0;
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    if (myId >= num_chunks) { // Use '>=' to ensure correct range checking
        return;
    }

    int text_start = myId * chunk_size; // Calculate start for this thread
    int text_length = text_start + chunk_size + pat_len - 1;

    // Use local register variable for text index
    int i = text_start + pat_len - 1;
    int k = 0;
    while (i < text_length && i < text_size) { // Combine boundary checks
        k = 0;
        while (k <= pat_len - 1 && pattern[pat_len - 1 - k] == text[i - k]) {
            k++;
        }
        if (k == pat_len) {
            ++count;
            i++; // Increment pointer by 1
        } else {
            i = i + shift_table[text[i]];
        }
    }

    // Store counts in shared memory
    s[threadIdx.x] = count;
    __syncthreads();

    // Use thread 0 to perform block-wide reduction
    if (threadIdx.x == 0) {
        int sum = 0;
        for (int idx = 0; idx < blockDim.x; idx++) { // Use blockDim for loop bound
            sum += s[idx];
        }
        d_output[blockIdx.x] = sum;
    }
}