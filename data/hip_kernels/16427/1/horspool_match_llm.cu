#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

#define NUM_THREADS_PER_BLOCK 512

int* create_shifts (char* pattern);

int linear_horspool_match (char* text, char* pattern, int* shift_table, unsigned int* num_matches, int chunk_size,
int num_chunks, int text_size, int pat_len, int myId);

// Optimized global kernel function
__global__ void horspool_match (char* text, char* pattern, int* shift_table, unsigned int* num_matches, int chunk_size, int num_chunks, int text_size, int pat_len) {
    const int TABLE_SIZ = 126;

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    if(myId >= num_chunks) { // Check for valid thread
        return;
    }

    int count = 0;
    int text_start = myId * chunk_size;
    int text_end = text_start + chunk_size + pat_len - 1;

    int i = text_start + pat_len - 1; // Initialize index

    while(i < text_end && i < text_size) {
        int k = 0;
        if (text[i] >= TABLE_SIZ || text[i] < 0) {
            // Skip unknown character
            ++i;
            continue;
        }
        
        while(k < pat_len && pattern[pat_len - 1 - k] == text[i - k]) {
            ++k; // Count matched characters
        }

        if(k == pat_len) {
            ++count; // Pattern found
            ++i; // Move to next character
        } else {
            // Shift index using table
            i += shift_table[text[i]];
        }
    }

    atomicAdd(num_matches, count);
}