#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addValue(int *array_val, int *b_array_val) {
    int x = threadIdx.x + blockIdx.x * blockDim.x; // Grid-stride loop for larger inputs
    int sum = 0;

    // Utilize shared memory for reducing global memory access time
    __shared__ int shared_array[COLUMNS];
    
    for (unsigned int i = 0; i < ROWS; i++) {
        if (x < COLUMNS) { // Ensure x is within bounds
            shared_array[x] = array_val[i * COLUMNS + x];
            __syncthreads(); // Synchronize threads to ensure shared memory is set
            sum += shared_array[x];
            __syncthreads(); // Synchronize threads before reusing shared memory
        }
    }

    if (x < COLUMNS) { // Ensure x is within bounds
        b_array_val[x] = sum;
    }
}