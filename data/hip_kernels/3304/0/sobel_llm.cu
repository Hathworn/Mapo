#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define threadsPerBlock 22

__device__ int pixelIndex(int x, int y, int width) {
    return (y * width + x);
}

__global__ void sobel(char* returnPixels, int width, char* pixels) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Check bounds to prevent access outside image boundaries
    if (x >= 1 && x < width - 1 && y >= 1 && y < width - 1) {
        int x00 = -1, x20 = 1;
        int x01 = -2, x21 = 2;
        int x02 = -1, x22 = 1;

        int y00 = -1, y10 = -2, y20 = -1;
        int y02 = 1, y12 = 2, y22 = 1;

        // Reduce repetitive calculations by prefetching values into variables
        int xm1_ym1 = pixels[pixelIndex(x - 1, y - 1, width)];
        int xm1_y = pixels[pixelIndex(x - 1, y, width)];
        int xm1_yp1 = pixels[pixelIndex(x - 1, y + 1, width)];
        int x_y = pixels[pixelIndex(x, y, width)];
        int x_yp1 = pixels[pixelIndex(x, y + 1, width)];
        int xp1_y = pixels[pixelIndex(x + 1, y, width)];
        int xp1_ym1 = pixels[pixelIndex(x + 1, y - 1, width)];
        int xp1_yp1 = pixels[pixelIndex(x + 1, y + 1, width)];

        int px = x00 * xm1_ym1 + x01 * xm1_y + x02 * xm1_yp1 + x20 * xp1_ym1 + x21 * xp1_y + x22 * xp1_yp1;
        int py = y00 * xm1_ym1 + y10 * x_y + y20 * xp1_ym1 + y02 * xm1_yp1 + y12 * x_yp1 + y22 * xp1_yp1;

        returnPixels[pixelIndex(x, y, width)] = sqrtf(float(px * px + py * py));
    }
}