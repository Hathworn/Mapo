#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addValue(int *array_val, int *b_array_val) {
    extern __shared__ int shared_array[];
    int cacheIndex = threadIdx.x;
    int globalIndex = blockIdx.x * COLUMNS + cacheIndex;
    
    // Load data into shared memory
    shared_array[cacheIndex] = array_val[globalIndex];
    __syncthreads();
    
    int i = blockDim.x / 2;
    while (i > 0) {
        if (cacheIndex < i) {
            // Reduce using shared memory
            shared_array[cacheIndex] += shared_array[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }
    
    // Write result from shared memory to global memory
    if (cacheIndex == 0) {
        b_array_val[blockIdx.x] = shared_array[0];
    }
}