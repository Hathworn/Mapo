#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findLowest(int numMin, int *array_val, int *cudaResult) {
    int tid = threadIdx.x;
    int low = tid * numMin;
    int high = min(low + numMin, blockDim.x * numMin); // Ensure the high bound doesn't exceed the array bounds
    int min = array_val[low];

    // Optimize by using a single loop with a condition
    for (unsigned int i = low + 1; i < high; i++) {
        if (array_val[i] < min) {
            min = array_val[i];
        }
    }

    cudaResult[tid] = min;
    __syncthreads(); // Optimization: ensures all threads write before anyone reads the result

    printf("Thread %d returned: %d \n", tid, min);
}