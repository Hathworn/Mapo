#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x;
    ii += threadIdx.y * (blockDim.x * gridDim.x);
    
    // Exit if the thread index exceeds the number of elements
    if (ii >= no_elements) return;

    // Use registers for storing scale_factor squared to avoid repeated calculations
    int scale_factor_sq = scale_factor * scale_factor;
    
    // Calculate the starting index for input data
    int ipidx_base = translate_idx_inv(ii, d1, d2, d3, scale_factor, 0, 0);

    // Initialize the temporary storage outside the loop
    float temp_sum = 0.0f;

    // Loop unrolling for better performance
    #pragma unroll
    for (int i = 0; i < scale_factor; i++) {
        #pragma unroll
        for (int j = 0; j < scale_factor; j++) {
            int ipidx = ipidx_base + i * d3 * scale_factor + j;
            temp_sum += gradOutput_data[ipidx];
        }
    }

    // Assign the result back to the global memory
    gradInput_data[ii] = temp_sum / scale_factor_sq;
}