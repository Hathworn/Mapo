#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SOFTMAX_NUM_THREADS 256

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    extern __shared__ float reduction_buffer[]; // Use dynamic shared memory

    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;

    const int idx = threadIdx.x;
    float tmp = 0;

    // One-level reduction to compute the inner products.
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Use parallel reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    tmp = reduction_buffer[0]; // The final sum result

    // Compute gradient
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}