#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void calcPrimes(int *d_IL, int *d_PL, int numOfPrimes, int lenInputList) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if(index < numOfPrimes) {
        int prime = d_PL[index];  // Load prime value once from global memory
        for(int i = prime * prime; i < lenInputList; i += prime) {  // Start from square of prime, increment by prime
            if(d_IL[i] != 0) {  // Avoid redundant zeroing
                d_IL[i] = 0;
            }
        }
    }
}