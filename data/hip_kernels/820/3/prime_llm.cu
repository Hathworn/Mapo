#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prime(int *a, int *b, int *c, int vector_size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride for better utilization of threads
    int stride = blockDim.x * gridDim.x;

    // Continue processing until all elements are handled
    for (int i = tid; i < vector_size; i += stride) {
        c[i] = a[i] + b[i];
    }
}