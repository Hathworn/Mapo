#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculation(char *a, char *b, int *c, int constant, int vector_size) {

    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Ensure thread is within bounds
    if (tid >= vector_size) return; 

    // Use shared memory for neighboring elements
    __shared__ char shared_a[blockDim.x + 2]; 

    // Load elements into shared memory
    shared_a[threadIdx.x + 1] = a[tid];

    if (threadIdx.x == 0) {
        shared_a[0] = a[tid > 0 ? tid - 1 : vector_size - 1]; // Load left neighbor
    }
    if (threadIdx.x == blockDim.x - 1 || tid == vector_size - 1) {
        shared_a[threadIdx.x + 2] = a[tid < vector_size - 1 ? tid + 1 : 0]; // Load right neighbor
    }

    __syncthreads(); // Synchronize to ensure all loads are complete

    // Do computation using shared memory
    int output_c = (shared_a[threadIdx.x] - shared_a[threadIdx.x + 2]) * b[tid] + shared_a[threadIdx.x + 1] * constant;

    // Write result
    c[tid] = output_c;
}