#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_calculation(char* dev_a, char* dev_b, char* dev_c, int k, int j, int num_matrices, int matrix_size) {
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate the offset in the data array for the matrix
    int offset = k * matrix_size * matrix_size + j * matrix_size;
    
    if (i < matrix_size) { // Ensure thread operates within valid range
        int index = offset + i; // Pre-compute index once
        dev_c[index] = dev_a[index] + dev_b[index]; // Direct addition to target
    }
}