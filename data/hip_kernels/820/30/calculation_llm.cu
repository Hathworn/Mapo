#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PADDED_BLOCK_SIZE (block_size + 2) // Define macro for padded block size

__global__ void calculation(int *a, int *b, int *c, int constant, int vector_size) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x; // Compute global thread ID

    __shared__ int sharedDataA[PADDED_BLOCK_SIZE]; // Shared memory with padding

    // Calculate left and right neighbor indices with wrap-around
    int leftNeighbor = (tid > 0) ? tid - 1 : vector_size - 1;
    int rightNeighbor = (tid < vector_size - 1) ? tid + 1 : 0;

    // Load data into shared memory
    if (tid < vector_size) {
        sharedDataA[threadIdx.x + 1] = a[tid]; // Center
        if (threadIdx.x == 0) {
            sharedDataA[0] = a[leftNeighbor]; // Left border
        }
        if (threadIdx.x == block_size - 1) {
            sharedDataA[PADDED_BLOCK_SIZE - 1] = a[rightNeighbor]; // Right border
        }
    }

    // Load B buffer (no shared memory needed)
    int curr_b = (tid < vector_size) ? b[tid] : 0;

    __syncthreads();

    // Perform calculation
    if (tid < vector_size) {
        int output_c = (sharedDataA[threadIdx.x] - sharedDataA[threadIdx.x + 2]) * curr_b;
        output_c += sharedDataA[threadIdx.x + 1] * constant;

        // Write result to global memory
        c[tid] = output_c;
    }
}