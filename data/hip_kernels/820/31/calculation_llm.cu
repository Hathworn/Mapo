#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define block_size 256  // Define block size as needed

__global__ void calculation(char *a, char *b, int *c, int constant, int vector_size) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;  // Calculate global thread id

    __shared__ char sharedDataA[block_size + 2]; // Shared memory with padding for borders
    char curr_b;

    // Use padding to avoid conditional statements for shared memory borders
    int sharedIdx = threadIdx.x + 1;
    sharedDataA[sharedIdx] = (tid < vector_size) ? a[tid] : 0;
    if (threadIdx.x == 0) {
        sharedDataA[0] = a[(tid > 0) ? (tid - 1) : (vector_size - 1)];
    }
    if (threadIdx.x == block_size - 1) {
        sharedDataA[block_size + 1] = a[(tid < vector_size - 1) ? (tid + 1) : 0];
    }

    curr_b = (tid < vector_size) ? b[tid] : 0;

    __syncthreads();

    if (tid < vector_size) {
        int output_c = (sharedDataA[sharedIdx - 1] - sharedDataA[sharedIdx + 1]) * curr_b;  // Calculate using neighbors
        output_c += sharedDataA[sharedIdx] * constant;
        c[tid] = output_c;  // Store result
    }
}