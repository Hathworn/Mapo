#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add( int *a, int *b, int *c, int vector_size ) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID

    if (tid < vector_size) {
        c[tid] = a[tid] + b[tid]; // Perform addition if within bounds
    }
}