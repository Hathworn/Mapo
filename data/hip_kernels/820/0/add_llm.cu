#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define block_size   32
#define vector_size  10000000

__global__ void add( int *a, int *b, int *c ) {
    int tid = (blockIdx.x*blockDim.x) + threadIdx.x;

    // Use a loop to handle multiple elements per thread.
    for (int i = tid; i < vector_size; i += blockDim.x * gridDim.x) {
        // Perform addition leveraging memory coalescing if applicable.
        c[i] = a[i] + b[i];
    }
}