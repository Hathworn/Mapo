#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculation(char* dev_a, char* dev_b, char* dev_c, int num_matrices, int matrix_size) {
    // Each thread handles a matrix
    int k = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (k >= num_matrices) return;

    // Use shared memory for matrix operations within a block
    extern __shared__ char shared_mem[];
    char* shared_a = shared_mem;
    char* shared_b = shared_a + matrix_size * matrix_size;

    for (int j = 0; j < matrix_size; j++) {
        for (int i = 0; i < matrix_size; i++) {
            int index = k * matrix_size * matrix_size + j * matrix_size + i;
            // Load data into shared memory to reduce global memory access
            shared_a[j * matrix_size + i] = dev_a[index];
            shared_b[j * matrix_size + i] = dev_b[index];
        }
    }

    // Synchronize threads to ensure all data is loaded
    __syncthreads();

    // If first element is different than 0, do the computation
    if (shared_a[0] != 0) {
        for (int j = 0; j < matrix_size; j++) {
            // If first value in the row of the matrix, do addition
            if (shared_a[j * matrix_size] < threshold) {
                for (int i = 0; i < matrix_size; i++) {
                    int index = k * matrix_size * matrix_size + j * matrix_size + i;
                    dev_c[index] = shared_a[j * matrix_size + i] + shared_b[j * matrix_size + i];
                }
            } else { // Do subtraction
                for (int i = 0; i < matrix_size; i++) {
                    int index = k * matrix_size * matrix_size + j * matrix_size + i;
                    dev_c[index] = shared_a[j * matrix_size + i] - shared_b[j * matrix_size + i];
                }
            }
        }
    }
}