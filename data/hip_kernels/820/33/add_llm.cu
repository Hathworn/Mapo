#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c, int vector_size) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;  // Calculate global thread index

    if (tid < vector_size) {  // Ensure within bounds of the vectors
        c[tid] = a[tid] + b[tid];  // Perform addition
    }
}