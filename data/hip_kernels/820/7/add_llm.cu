#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check index bounds before performing addition
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}