#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sub_calculation(char* dev_a, char* dev_b, char* dev_c, int k, int j, int num_matrices, int matrix_size) {
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread handles valid matrix element index
    if (i < matrix_size) {
        int index = (k * matrix_size + j) * matrix_size + i;
        dev_c[index] = dev_a[index] - dev_b[index];
    }
}