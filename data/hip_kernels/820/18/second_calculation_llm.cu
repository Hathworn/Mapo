#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void second_calculation(char* dev_a, char* dev_b, char* dev_c, int k, int num_matrices, int matrix_size) {
    // Each thread handles a matrix
    int j = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (j >= matrix_size) return;

    // Precompute base index outside the loop for efficiency
    int base_index = k * matrix_size * matrix_size + j * matrix_size;
    bool add_operation = dev_a[base_index] < threshold;

    for (int i = 0; i < matrix_size; i++) {
        int index = base_index + i;
        if (add_operation) {
            dev_c[index] = dev_a[index] + dev_b[index]; // Perform addition
        } else {
            dev_c[index] = dev_a[index] - dev_b[index]; // Perform subtraction
        }
    }
}