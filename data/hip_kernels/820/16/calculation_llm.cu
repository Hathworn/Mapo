#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculation(char *a, char *b, int *c, int constant, int vector_size) {

    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    if (tid < vector_size) {
        // Use shared memory to minimize global memory access
        extern __shared__ char shared_a[];
        
        // Load data into shared memory with boundary wrapping
        shared_a[threadIdx.x] = a[tid];
        if (threadIdx.x == 0) {
            shared_a[blockDim.x] = a[tid < vector_size - 1 ? tid + 1 : 0];
            shared_a[-1] = a[tid > 0 ? tid - 1 : vector_size - 1];
        }
        __syncthreads();
        
        // Read in inputs
        char prev_a = shared_a[threadIdx.x - 1];
        char curr_a = shared_a[threadIdx.x];
        char post_a = shared_a[threadIdx.x + 1];
        char curr_b = b[tid];
        
        // Do computation
        int output_c = (prev_a - post_a) * curr_b + curr_a * constant;
        
        // Write result
        c[tid] = output_c;
    }
}