#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculation(char *a, char *b, int *c, int constant, int vector_size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; 

    if (tid < vector_size) {
        // Using shared memory to improve memory access efficiency
        extern __shared__ char shared_a[];

        // Load data into shared memory
        shared_a[threadIdx.x] = a[tid];
        if (threadIdx.x == 0) {
            shared_a[blockDim.x] = a[tid > 0 ? tid - 1 : (vector_size - 1)];
            shared_a[blockDim.x + 1] = a[tid < (vector_size - 1) ? tid + 1 : 0];
        }
        __syncthreads();

        // Compute indices for previous and post elements
        char prev_a = shared_a[threadIdx.x == 0 ? blockDim.x : threadIdx.x - 1];
        char curr_a = shared_a[threadIdx.x];
        char post_a = shared_a[threadIdx.x == blockDim.x - 1 ? blockDim.x + 1 : threadIdx.x + 1];

        char curr_b = b[tid];

        // Do computation
        int output_c = (prev_a - post_a) * curr_b + curr_a * constant;

        // Write result
        c[tid] = output_c;
    }
}