#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prime_generator(int *input, int *prime_list, int *total_input, int *seed) {
    // Using shared memory to reduce global memory access.
    __shared__ int shared_seed;
    if (threadIdx.x == 0) {
        shared_seed = seed[0];
    }
    __syncthreads();

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int primeno = prime_list[i];
    int total = shared_seed * shared_seed;

    // Using stride loop to optimize memory access in case grid size is smaller than total.
    for (int k = shared_seed + i; k < total; k += blockDim.x * gridDim.x) {
        if (k % primeno == 0) {
            input[k] = 1;
        }
    }
}