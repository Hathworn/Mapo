#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matMult(int* a, int* b, int* res, unsigned int rows, unsigned int k, unsigned int cols) {
    unsigned int r = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (r < rows && c < cols) {
        int sum = 0;  // Initialize sum inside the loop
        for (int x = 0; x < k; x++) {
            sum += a[r * k + x] * b[x * cols + c];  // Correct the operation from addition to multiplication
        }
        res[r * cols + c] = sum;
    }
}