#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculation(char *a, char *b, int *c, int constant, int vector_size) {

    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure valid thread index
    if (tid < vector_size) {

        // Efficient memory access using shared memory
        extern __shared__ char shared_a[];

        // Load shared data
        shared_a[threadIdx.x] = a[tid];
        
        // Handle boundary conditions
        char prev_a, post_a;

        if (threadIdx.x == 0) {
            prev_a = tid > 0 ? a[tid - 1] : a[vector_size - 1];
        } else {
            prev_a = shared_a[threadIdx.x - 1];
        }

        if (threadIdx.x == blockDim.x - 1 || tid == vector_size - 1) {
            post_a = tid < (vector_size - 1) ? a[tid + 1] : a[0];
        } else {
            post_a = shared_a[threadIdx.x + 1];
        }

        char curr_b = b[tid];

        // Perform computation with reduced arithmetic operations
        int output_c = (prev_a - post_a) * curr_b + shared_a[threadIdx.x] * constant;

        // Store the result
        c[tid] = output_c;
    }
}