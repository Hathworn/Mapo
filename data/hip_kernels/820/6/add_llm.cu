#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    int col = 10;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    int idx = i * col + j; // Calculate index once
    c[idx] = a[idx] + b[idx]; // Access memory efficiently
}