#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c, int vector_size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID
    // Ensure the thread processes valid data
    if (tid < vector_size) {
        c[tid] = a[tid] + b[tid]; // Perform vector addition
    }
}