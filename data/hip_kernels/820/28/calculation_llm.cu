#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculation(int *a, int *b, int *c, int constant, int vector_size) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (tid < vector_size) {
        // Calculate indices with modulo to avoid branching
        int prev_tid = (tid + vector_size - 1) % vector_size;
        int post_tid = (tid + 1) % vector_size;

        // Read in inputs
        int prev_a = a[prev_tid];
        int curr_a = a[tid];
        int post_a = a[post_tid];

        int curr_b = b[tid];
        
        // Do computation
        int output_c = (prev_a - post_a) * curr_b + curr_a * constant;

        // Write result
        c[tid] = output_c;
    }
}