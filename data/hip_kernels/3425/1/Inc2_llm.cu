#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc2(float *Ad, float *Bd) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx < LEN) {
        float temp = Ad[tx];  // Load value once
        for (int i = 0; i < ITER; i++) {
            temp += 1.0f;  // Accumulate result in register
            Ad[tx] = temp; // Write back to global memory
            Bd[tx] = temp; // Remove inner loop
        }
    }
}