#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    // Calculate the unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a stride loop for better utilization of GPU resources
    for (int i = idx; i < N; i += blockDim.x * gridDim.x) {
        a[i] = a[i] * a[i];
    }
}