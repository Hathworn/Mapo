#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];

    // Load input into shared memory
    int tid = threadIdx.x;
    int index = tid + blockIdx.x * blockDim.x;
    tmp[tid] = input[index];

    __syncthreads();

    // Unrolled reduction steps for better performance
    // Reduce within a single warp if possible
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            tmp[tid] += tmp[tid + s];
        }
        __syncthreads();
    }

    // Write result to global memory
    if (tid == 0) {
        output[blockIdx.x] = tmp[0];
    }
}