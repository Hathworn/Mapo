#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_add(float* a, float* b, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        a[i] += b[i]; // Use compound assignment for brevity
    }
}