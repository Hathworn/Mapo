#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bigstencil(int* in, int* out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to improve memory access speed
    __shared__ int sharedIn[256]; // Assuming blockDim.x is 256, adjust if different

    // Load input into shared memory
    sharedIn[threadIdx.x] = in[i];
    __syncthreads();

    // Perform computation using shared memory
    out[i] = sharedIn[threadIdx.x] + 2;
}