#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    // Use shared memory for input vector to reduce global memory accesses
    extern __shared__ double shared_vector_in[];
    
    // Load vector_in into shared memory
    if (threadIdx.x < dim_mn) {
        shared_vector_in[threadIdx.x] = vector_in[threadIdx.x];
    }
    __syncthreads(); // Ensure all threads have loaded their part of vector_in

    double out = 0.0;
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (i < dim_mn) {
        // Unroll loop for performance
        #pragma unroll
        for (long j = 0; j < dim_mn; j++) {
            out += matrix[i * dim_mn + j] * shared_vector_in[j];
        }
        vector_out[i] = out;
    }
}