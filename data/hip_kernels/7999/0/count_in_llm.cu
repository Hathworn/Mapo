#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

#define ULL unsigned long long

const long MAXDIM = 10;
const double RMIN = 2.0;
const double RMAX = 7.0;

#define MAX_THREADS 1024
#define MAX_BLOCKS 65535

//Optimized global kernel code that runs on the device
__global__ void count_in(ULL *dev_count, long dev_ntotal, long dev_ndim, long dev_halfb, double dev_rsquare, long dev_base) {
    //Calculate the position of this kernel in the data
    ULL blockID = (blockIdx.y * gridDim.x) + blockIdx.x;
    ULL pos = (blockID * blockDim.x) + threadIdx.x;

    //If this threads position in the data is further than we need to calculate
    //Then we return
    if (pos >= dev_ntotal) return;

    double rtestsq = 0;
    long idx = 0;
    long index[MAXDIM] = {0};  //Optimize: Initialize index array with zeros

    //Convert the decimal number into another base system
    ULL temp_pos = pos;  //Optimize: Use temporary variable to retain original position value
    while (temp_pos != 0) {
        long rem = temp_pos % dev_base;
        temp_pos = temp_pos / dev_base;
        index[idx] = rem;
        ++idx;
    }

    for (long k = 0; k < dev_ndim; ++k) {
        double xk = index[k] - dev_halfb;
        rtestsq += xk * xk;
    }

    //If the value is inside the sphere
    //Atomically add 1 to the count
    if (rtestsq < dev_rsquare) {
        atomicAdd(dev_count, 1);
    }
}