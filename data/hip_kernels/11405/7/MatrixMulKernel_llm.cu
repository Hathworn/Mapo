#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 16  // Define tile size

__global__ void MatrixMulKernel(float *d_x, float *d_y, float *d_z, int Width) {

    // Define shared memory for tiles
    __shared__ float tile_x[TILE_SIZE][TILE_SIZE];
    __shared__ float tile_y[TILE_SIZE][TILE_SIZE];

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    float kernelSum = 0;

    if (idx < Width && idy < Width) {
        for (int t = 0; t < Width / TILE_SIZE; ++t) {
            // Load tiles into shared memory
            tile_x[threadIdx.y][threadIdx.x] = d_x[idy * Width + (t * TILE_SIZE + threadIdx.x)];
            tile_y[threadIdx.y][threadIdx.x] = d_y[(t * TILE_SIZE + threadIdx.y) * Width + idx];
            __syncthreads();  // Synchronize to ensure complete tile load

            for (int k = 0; k < TILE_SIZE; ++k) {
                kernelSum += tile_x[threadIdx.y][k] * tile_y[k][threadIdx.x];
            }
            __syncthreads();  // Synchronize for next tile calculation
        }
        d_z[idy * Width + idx] = kernelSum;
    }
}