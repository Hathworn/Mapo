#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecProductKernel(float *d_z, const float *d_x, const float *d_y, unsigned int N)
{
    // Utilize shared memory for faster access
    extern __shared__ float shared_memory[];
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;

    if (idx < N) {
        // Load data into shared memory
        shared_memory[tid] = d_x[idx] * d_y[idx];
        __syncthreads();  // Ensure all loads are complete

        // Write the result back to global memory
        d_z[idx] = shared_memory[tid];
    }
}