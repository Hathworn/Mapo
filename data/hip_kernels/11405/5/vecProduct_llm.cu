#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecProduct(int *d_x, int *d_y, int *d_z, int N) {
    // Calculate the global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // The stride for grid-stride loop
    int stride = blockDim.x * gridDim.x;

    // Use grid-stride loop to enable processing of more elements and improve occupancy
    for (int i = idx; i < N; i += stride) {
        d_z[i] = d_x[i] * d_y[i];
    }
}