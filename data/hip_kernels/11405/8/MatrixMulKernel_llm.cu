#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulKernel(int *d_x, int *d_y, int *d_z, int Block_Width, int M, int N) {

    // Calculate row and column indexes
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize variable for summation
    int kernelSum = 0;

    // Check bounds
    if (row < N && col < N) {
        // Loop unrolling for faster computation; assumes Block_Width is a multiple of 4
        for (int i = 0; i < Block_Width; i += 4) {
            #pragma unroll
            for (int j = 0; j < 4; ++j) {
                kernelSum += d_x[col * Block_Width + (i + j)] * d_y[(i + j) * Block_Width + row];
            }
        }
        d_z[row * Block_Width + col] = kernelSum;
    }
}