#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Using shared memory to optimize memory access and reduce global memory traffic
__global__ void vecProductKernel(float *d_z, const float *d_x, const float *d_y, unsigned int N)
{
    extern __shared__ float shared_x[];
    extern __shared__ float shared_y[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (idx < N) {
        shared_x[threadIdx.x] = d_x[idx];
        shared_y[threadIdx.x] = d_y[idx];
    }
    __syncthreads();

    // Perform computation using shared memory
    if (idx < N) {
        d_z[idx] = shared_x[threadIdx.x] * shared_y[threadIdx.x];
    }
}