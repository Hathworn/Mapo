#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Thread 0 is responsible for dividing accuracy by N
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}