#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_hello() {
    // Use grid and block size to calculate the unique thread index for better distribution
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello World from GPU! %d\n", idx);
}
```
