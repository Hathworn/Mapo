#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t

// == Dimension rearrangement Kernel

__global__ void CorrelateDataBackward1Subtract_1d(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels, int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2, int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size, const float *bottom0, const float *bottom1, float *bottom1diff, const float *topdiff)
{
    // Use cooperative indexing to eliminate shared memory bank conflicts
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= nthreads) return; // Exits threads outside this range immediately

    int l = index % bottomwidth + pad_size; // w-pos
    int m = (index / bottomwidth) % bottomheight; // h-pos
    int n = (index / bottomwidth / bottomheight) % bottomchannels; // channels

    // Pre-compute constants used in the loop for better performance
    const int round_off_s1 = stride1 * ROUND_OFF;
    const int round_off = ROUND_OFF;
    
    float sum = 0;
    for(int o = x_shift; o < x_shift + neighborhood_grid_width; o++) {
        int s2o = stride2 * o;

        // Precompute the range boundaries using shared constants
        int xmin = (l - 2 * kernel_radius - max_displacement - s2o + round_off_s1 - 1) / stride1 + 1 - round_off;
        int ymin = (m - 2 * kernel_radius + round_off_s1 - 1) / stride1 + 1 - round_off;
        int xmax = (l - max_displacement - s2o + round_off_s1) / stride1 - round_off;
        int ymax = (m + round_off_s1) / stride1 - round_off;

        if(xmax >= 0 && ymax >= 0 && xmin <= topwidth - 1 && ymin <= topheight - 1) {
            xmin = max(0, xmin);
            xmax = min(topwidth - 1, xmax);
            ymin = max(0, ymin);
            ymax = min(topheight - 1, ymax);

            int idxbot = ((item * pbottomheight + m) * pbottomwidth + (l - s2o)) * bottomchannels + n;
            float bot0tmp = bottom0[idxbot];
            float bot1tmp = bottom1[idxbot];
            float sign = (bot0tmp >= bot1tmp) ? float(-1.0) : float(1.0);

            int op = (o - x_shift);
            int idxOpOffset = (item * topchannels + op);

            for(int y = ymin; y <= ymax; y++) {
                for(int x = xmin; x <= xmax; x++) {
                    int idxtopdiff = (idxOpOffset * topheight + y) * topwidth + x;
                    sum += topdiff[idxtopdiff] * sign;
                }
            }
        }
    }
    const int sumelems = (kernel_radius * 2 + 1) * (kernel_radius * 2 + 1) * bottomchannels;
    bottom1diff[index + item * bottomcount] = sum / (float)sumelems;
}