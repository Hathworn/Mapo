#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t

// == Dimension rearrangement Kernel

__global__ void CorrelateDataBackward0Subtract_1d(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels, int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2, int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size, float *bottom0diff, const float *bottom0, const float *bottom1, const float *topdiff)
{
    CUDA_KERNEL_LOOP(index, nthreads) {
        int l = index % bottomwidth + pad_size; //w-pos
        int m = (index / bottomwidth) % bottomheight; //h-pos
        int n = (index / bottomwidth / bottomheight) % bottomchannels; //channels

        //Get X,Y ranges and clamp
        const int round_off = ROUND_OFF;
        const int round_off_s1 = stride1 * round_off;

        int xmin = (l - 2*kernel_radius - max_displacement + round_off_s1 - 1) / stride1 + 1 - round_off;
        int ymin = (m - 2*kernel_radius + round_off_s1 - 1) / stride1 + 1 - round_off;

        int xmax = (l - max_displacement + round_off_s1) / stride1 - round_off;
        int ymax = (m + round_off_s1) / stride1 - round_off;

        float sum = 0;
        if(xmax >= 0 && ymax >= 0 && xmin <= topwidth-1 && ymin <= topheight-1)
        {
            xmin = max(0, xmin);
            xmax = min(topwidth-1, xmax);
            ymin = max(0, ymin);
            ymax = min(topheight-1, ymax);

            // Precompute index and inner loop offset
            int offset_n = item * pbottomheight * pbottomwidth * bottomchannels + n;
            int offset_top = item * topchannels * topheight * topwidth;

            for(int o = x_shift; o < x_shift + neighborhood_grid_width; o++) {
                int s2o = stride2 * o;
                int idxbot_base = offset_n + (m * pbottomwidth + (l + s2o)) * bottomchannels;

                float sign = (bottom0[idxbot_base] >= bottom1[idxbot_base]) ? 1.0f : -1.0f;
                int op = o - x_shift;
                int idxopoffset = op * topheight * topwidth + offset_top;

                for(int y = ymin; y <= ymax; y++) {
                    int idxy = y * topwidth;
                    for(int x = xmin; x <= xmax; x++) {
                        int idxtopdiff = idxopoffset + idxy + x; 
                        sum += topdiff[idxtopdiff] * sign;
                    }
                }
            }
        }
        const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
        bottom0diff[index + item*bottomcount] = sum / float(sumelems);
    }
}