#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t

// == Dimension rearrangement Kernel

__global__ void CorrelateData_1d(const int nthreads, int num, int topwidth, int topheight, int topchannels, int topcount, int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int kernel_size, int stride1, int stride2, int bottomwidth, int bottomheight, int bottomchannels, const float *bottom0, const float *bottom1, float *top)
{
    extern __shared__ char patch_data_char[];

    float *patch_data = (float *)patch_data_char;

    // First (upper left) position of kernel upper-left corner in current center position of neighborhood in image 1
    int x1 = blockIdx.x * stride1 + max_displacement;
    int y1 = blockIdx.y * stride1;
    int item = blockIdx.z;
    int ch_off = threadIdx.x;

    // Load 3D patch into shared memory
    for(int j = 0; j < kernel_size; j++) { // HEIGHT
        for(int i = 0; i < kernel_size; i++) { // WIDTH
            int ji_off = ((j * kernel_size) + i) * bottomchannels;
            for(int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK * THREADS_PER_WARP)) { // CHANNELS
                int idx1 = ((item * bottomheight + y1 + j) * bottomwidth + x1 + i) * bottomchannels + ch;
                int idxPatchData = ji_off + ch;
                patch_data[idxPatchData] = bottom0[idx1];
            }
        }
    }

    __syncthreads();

    __shared__ float sum[WARPS_PER_BLOCK * THREADS_PER_WARP];

    // Compute correlation
    for(int top_channel = 0; top_channel < topchannels; top_channel++) {
        sum[ch_off] = 0;
        
        int s2o = (top_channel % neighborhood_grid_width + x_shift) * stride2;
        int x2 = x1 + s2o; // Move x2 calculation outside loops for reuse

        for(int j = 0; j < kernel_size; j++) { // HEIGHT
            for(int i = 0; i < kernel_size; i++) { // WIDTH
                int ji_off = ((j * kernel_size) + i) * bottomchannels;
                for(int ch = ch_off; ch < bottomchannels; ch += (WARPS_PER_BLOCK * THREADS_PER_WARP)) { // CHANNELS
                    int idxPatchData = ji_off + ch;
                    int idx2 = ((item * bottomheight + y1 + j) * bottomwidth + x2 + i) * bottomchannels + ch;
                    
                    sum[ch_off] += patch_data[idxPatchData] * bottom1[idx2];
                }
            }
        }

        __syncthreads();

        if (ch_off == 0) {
            float total_sum = 0;
            for(int idx = 0; idx < WARPS_PER_BLOCK * THREADS_PER_WARP; idx++) {
                total_sum += sum[idx];
            }
            const int sumelems = kernel_size * kernel_size * bottomchannels;
            const int index = ((top_channel * topheight + blockIdx.y) * topwidth) + blockIdx.x;
            top[index + item * topcount] = total_sum / (float)sumelems;
        }
    }
}