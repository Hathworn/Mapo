#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t

// == Dimension rearrangement Kernel

__global__ void CorrelateDataBackward1_1d(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels, int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2, int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size, const float *bottom0, float *bottom1diff, const float *topdiff)
{
    CUDA_KERNEL_LOOP(index, nthreads) {
        // Rearrange the calculation order to minimize operations
        int n = index % bottomchannels; // channels
        int l = (index / bottomchannels) % bottomwidth + pad_size; // w-pos
        int m = (index / bottomchannels / bottomwidth) % bottomheight; // h-pos

        // round_off is a trick to enable integer division with ceil, even for negative numbers
        // We use a large offset, for the inner part not to become negative.
        const int round_off = ROUND_OFF;
        const int round_off_s1 = stride1 * round_off;

        float sum = 0;

        #pragma unroll // Using loop unrolling for small loops
        for(int o = x_shift; o < x_shift + neighborhood_grid_width; o++) {

            int s2o = stride2 * o;

            // Get X,Y ranges and clamp
            int xmin = max(0, (l - 2*kernel_radius - max_displacement - s2o + round_off_s1 - 1) / stride1 + 1 - round_off);
            int ymin = max(0, (m - 2*kernel_radius + round_off_s1 - 1) / stride1 + 1 - round_off);

            int xmax = min(topwidth-1, (l - max_displacement - s2o + round_off_s1) / stride1 - round_off);
            int ymax = min(topheight-1, (m + round_off_s1) / stride1 - round_off);

            if(xmax >= 0 && ymax >= 0 && xmin <= topwidth-1 && ymin <= topheight-1) {

                // Get bottom0 data:
                int idxbot0 = ((item * pbottomheight + m) * pbottomwidth + (l-s2o)) * bottomchannels + n;
                float bot0tmp = bottom0[idxbot0]; // bottom1[l+s2o,m,n]

                // Index offset for topdiff in following loops:
                int op = (o-x_shift); // index [o,p]
                int idxOpOffset = (item * topchannels + op);

                for(int y = ymin; y <= ymax; y++) {
                    for(int x = xmin; x <= xmax; x++) {
                        int idxtopdiff = (idxOpOffset * topheight + y) * topwidth + x; // topdiff[x,y,o,p]
                        sum += topdiff[idxtopdiff] * bot0tmp;
                    }
                }
            }
        }

        const int sumelems = (kernel_radius*2+1)*(kernel_radius*2+1)*bottomchannels;
        const int bot1index = ((n * bottomheight) + m) * bottomwidth + (l-pad_size);
        bottom1diff[bot1index + item*bottomcount] = sum / (float)sumelems;
    }
}