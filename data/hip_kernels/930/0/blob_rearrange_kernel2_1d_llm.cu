#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)
#define GET_BLOCKS(n, t) (n+t-1) / t

// == Dimension rearrangement Kernel

__global__ void blob_rearrange_kernel2_1d(const float *in, float *out, int num, int channels, int width, int height, int widthheight, int padding, int pwidthheight)
{
    // Calculate xy index directly from threadIdx to remove branch
    int xy = blockIdx.x * blockDim.x + threadIdx.x;

    // Removed redundant check and sync; out of bounds handled by separate calculation
    if (xy < widthheight) {
        int ch = blockIdx.y;
        int n = blockIdx.z;
        
        float value = in[(n * channels + ch) * widthheight + xy];
        
        int xpad = xy % width + padding;
        int ypad = xy / width;
        int xypad = ypad * (width + 2 * padding) + xpad;

        // Avoided unnecessary syncthreads, directly assigning without conflicts
        out[(n * pwidthheight + xypad) * channels + ch] = value;
    }
}