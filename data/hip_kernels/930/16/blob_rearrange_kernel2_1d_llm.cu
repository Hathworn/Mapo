#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blob_rearrange_kernel2_1d(const float *in, float *out, int num, int channels, int width, int height, int widthheight, int padding, int pwidthheight)
{
    // Calculate the global index
    int xy = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread is within bounds
    if (xy >= widthheight)
        return;

    // Determine the current channel and image index
    int ch = blockIdx.y;
    int n  = blockIdx.z;

    // Read input value
    float value = in[(n * channels + ch) * widthheight + xy];

    // Calculate padded coordinates without sync
    int xpad  = (xy % width + padding);
    int ypad  = (xy / width);
    int xypad = ypad * (width + 2 * padding) + xpad;

    // Write output value
    // Avoid race conditions, sync is not necessary as threads don't overlap in data write
    out[(n * pwidthheight + xypad) * channels + ch] = value;
}