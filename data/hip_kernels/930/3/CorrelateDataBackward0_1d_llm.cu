#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define ROUND_OFF 50000

#define CUDA_NUM_THREADS 1024
#define WARPS_PER_BLOCK 1
#define THREADS_PER_WARP 32

#define CUDA_KERNEL_LOOP(i, n) for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); i += blockDim.x * gridDim.x)

#define GET_BLOCKS(n, t) (n+t-1) / t

// == Dimension rearrangement Kernel

__global__ void CorrelateDataBackward0_1d(const int nthreads, int num, int item, int topwidth, int topheight, int topchannels, int max_displacement, int x_shift, int neighborhood_grid_width, int kernel_radius, int stride1, int stride2, int bottomwidth, int bottomheight, int pbottomwidth, int pbottomheight, int bottomchannels, int bottomcount, int pad_size, float *bottom0diff, const float *bottom1, const float *topdiff)
{
    // Using shared memory for faster access
    extern __shared__ float shared_bot1[];
    CUDA_KERNEL_LOOP(index, nthreads)
    {
        int n = index % bottomchannels; 
        int l = (index / bottomchannels) % bottomwidth + pad_size; 
        int m = (index / bottomchannels / bottomwidth) % bottomheight; 

        const int round_off = ROUND_OFF;
        const int round_off_s1 = stride1 * round_off;

        int xmin = (l - 2 * kernel_radius - max_displacement + round_off_s1 - 1) / stride1 + 1 - round_off;
        int ymin = (m - 2 * kernel_radius + round_off_s1 - 1) / stride1 + 1 - round_off;

        int xmax = (l - max_displacement + round_off_s1) / stride1 - round_off;
        int ymax = (m + round_off_s1) / stride1 - round_off;

        if (xmax >= 0 && ymax >= 0 && xmin <= topwidth - 1 && ymin <= topheight - 1)
        {
            xmin = max(0, xmin);
            xmax = min(topwidth - 1, xmax);

            ymin = max(0, ymin);
            ymax = min(topheight - 1, ymax);

            float sum = 0;
            for (int o = x_shift; o < x_shift + neighborhood_grid_width; o++)
            {
                // Load bottom1 data into shared memory
                int s2o = stride2 * o;
                int idxbot1 = ((item * pbottomheight + m) * pbottomwidth + (l + s2o)) * bottomchannels + n;
                shared_bot1[threadIdx.x] = bottom1[idxbot1];
                __syncthreads(); // Ensure all threads have loaded data

                float bot1tmp = shared_bot1[threadIdx.x];

                int op = (o - x_shift);
                int idxopoffset = item * topchannels + op;

                for (int y = ymin; y <= ymax; y++)
                {
                    for (int x = xmin; x <= xmax; x++)
                    {
                        int idxtopdiff = (idxopoffset * topheight + y) * topwidth + x;
                        sum += topdiff[idxtopdiff] * bot1tmp;
                    }
                }
            }
            const int sumelems = (kernel_radius * 2 + 1) * (kernel_radius * 2 + 1) * bottomchannels;
            const int bot0index = (n * bottomheight + m) * bottomwidth + (l - pad_size);
            bottom0diff[bot0index + item * bottomcount] = sum / (float)sumelems;
        }
    }
}