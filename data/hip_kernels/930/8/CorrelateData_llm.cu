#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CorrelateData(const int nthreads, int num, int topwidth, int topheight, int topchannels, int topcount, int max_displacement, int neighborhood_grid_radius, int neighborhood_grid_width, int kernel_radius, int kernel_size, int stride1, int stride2, int bottomwidth, int bottomheight, int bottomchannels, const float *bottom0, const float *bottom1, float *top)
{
    extern __shared__ float patch_data[];

    // First (upper left) position of kernel upper-left corner in current center position of neighborhood in image 1
    int x1 = blockIdx.x * stride1 + max_displacement;
    int y1 = blockIdx.y * stride1 + max_displacement;
    int item = blockIdx.z;
    int ch_off = threadIdx.x;

    // Load 3D patch into shared memory
    for (int j = 0; j < kernel_size; j++) { // HEIGHT
        for (int i = 0; i < kernel_size; i++) { // WIDTH
            int ji_off = ((j * kernel_size) + i) * bottomchannels;
            for (int ch = ch_off; ch < bottomchannels; ch += blockDim.x) { // CHANNELS
                int idx1 = ((item * bottomheight + y1+j) * bottomwidth + x1+i) * bottomchannels + ch;
                int idxPatchData = ji_off + ch;
                patch_data[idxPatchData] = bottom0[idx1];
            }
        }
    }

    __syncthreads();

    float sum = 0.0f; // Use register memory for sum

    // Compute correlation
    for (int top_channel = 0; top_channel < topchannels; top_channel++) {
        int s2o = (top_channel % neighborhood_grid_width - neighborhood_grid_radius) * stride2;
        int s2p = (top_channel / neighborhood_grid_width - neighborhood_grid_radius) * stride2;

        for (int j = 0; j < kernel_size; j++) { // HEIGHT
            for (int i = 0; i < kernel_size; i++) { // WIDTH
                int ji_off = ((j * kernel_size) + i) * bottomchannels;
                for (int ch = ch_off; ch < bottomchannels; ch += blockDim.x) { // CHANNELS
                    int x2 = x1 + s2o;
                    int y2 = y1 + s2p;

                    int idxPatchData = ji_off + ch;
                    int idx2 = ((item * bottomheight + y2+j) * bottomwidth + x2+i) * bottomchannels + ch;

                    sum += patch_data[idxPatchData] * bottom1[idx2];
                }
            }
        }

        __syncthreads();

        // Use atomicAdd to ensure sum aggregation
        if (ch_off == 0) {
            const int sumelems = kernel_size * kernel_size * bottomchannels;
            const int index = ((top_channel * topheight + blockIdx.y) * topwidth) + blockIdx.x;
            atomicAdd(&top[index + item * topcount], sum / (float)sumelems);
        }
    }
}