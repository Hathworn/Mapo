#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blob_rearrange_kernel2(const float *in, float *out, int num, int channels, int width, int height, int widthheight, int padding, int pwidthheight)
{
    // Calculate global thread index
    int xy = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread is within bounds
    if (xy >= widthheight)
        return;

    // Calculate channel and batch index
    int ch = blockIdx.y;
    int n  = blockIdx.z;

    // Optimize memory access by directly accessing input and output data
    float value = in[(n * channels + ch) * widthheight + xy];

    // Pre-compute padded indices to avoid repeated computation
    int xpad  = (xy % width) + padding;
    int ypad  = (xy / width) + padding;
    int xypad = ypad * (width + 2 * padding) + xpad;

    out[(n * pwidthheight + xypad) * channels + ch] = value;
}
```
