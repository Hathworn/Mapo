#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SoftClipKernel(const float* p_Input, float* p_Output, int p_Width, int p_Height, 
                               float p_SoftClipA, float p_SoftClipB, float p_SoftClipC, float p_SoftClipD, 
                               float p_SoftClipE, float p_SoftClipF, int p_SwitchA, int p_SwitchB, int p_Source) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < p_Width && y < p_Height) {
        const int index = (y * p_Width + x) * 4;
        float r = p_Input[index];
        float g = p_Input[index + 1];
        float b = p_Input[index + 2];

        float cr = (powf(10.0f, (1023.0f * r - 685.0f) / 300.0f) - 0.0108f) / (1.0f - 0.0108f);
        float cg = (powf(10.0f, (1023.0f * g - 685.0f) / 300.0f) - 0.0108f) / (1.0f - 0.0108f);
        float cb = (powf(10.0f, (1023.0f * b - 685.0f) / 300.0f) - 0.0108f) / (1.0f - 0.0108f);

        float lr = r > 0.1496582f ? (powf(10.0f, (r - 0.385537f) / 0.2471896f) - 0.052272f) / 5.555556f 
                                  : (r - 0.092809f) / 5.367655f;
        float lg = g > 0.1496582f ? (powf(10.0f, (g - 0.385537f) / 0.2471896f) - 0.052272f) / 5.555556f 
                                  : (g - 0.092809f) / 5.367655f;
        float lb = b > 0.1496582f ? (powf(10.0f, (b - 0.385537f) / 0.2471896f) - 0.052272f) / 5.555556f 
                                  : (b - 0.092809f) / 5.367655f;

        float mr = lr * 1.617523f + lg * -0.537287f + lb * -0.080237f;
        float mg = lr * -0.070573f + lg * 1.334613f + lb * -0.26404f;
        float mb = lr * -0.021102f + lg * -0.226954f + lb * 1.248056f;

        // Determine the source based on the p_Source parameter
        float sr = (p_Source == 0) ? r : (p_Source == 1) ? cr : mr;
        float sg = (p_Source == 0) ? g : (p_Source == 1) ? cg : mg;
        float sb = (p_Source == 0) ? b : (p_Source == 1) ? cb : mb;

        float Lr = fminf(sr, 1.0f);
        float Lg = fminf(sg, 1.0f);
        float Lb = fminf(sb, 1.0f);

        float Hr = fmaxf(sr, 1.0f) - 1.0f;
        float Hg = fmaxf(sg, 1.0f) - 1.0f;
        float Hb = fmaxf(sb, 1.0f) - 1.0f;

        float rr = p_SoftClipA;
        float gg = p_SoftClipB;
        float aa = p_SoftClipC;
        float bb = p_SoftClipD;
        float ss = 1.0f - (p_SoftClipE / 10.0f);
        float sf = 1.0f - p_SoftClipF;

        float Hrr = Hr * powf(2.0f, rr);
        float Hgg = Hg * powf(2.0f, rr);
        float Hbb = Hb * powf(2.0f, rr);

        float HR = (Hrr <= 1.0f) ? 1.0f - powf(1.0f - Hrr, gg) : Hrr;
        float HG = (Hgg <= 1.0f) ? 1.0f - powf(1.0f - Hgg, gg) : Hgg;
        float HB = (Hbb <= 1.0f) ? 1.0f - powf(1.0f - Hbb, gg) : Hbb;

        float R = Lr + HR;
        float G = Lg + HG;
        float B = Lb + HB;

        // Soft clipping
        float softr = aa == 1.0f ? R : (R > aa ? (-1.0f / ((R - aa) / (bb - aa) + 1.0f) + 1.0f) * (bb - aa) + aa : R);
        float softR = (bb == 1.0f) ? softr : (softr > 1.0f - (bb / 50.0f) ? (-1.0f / ((softr - (1.0f - (bb / 50.0f))) / 
                            (1.0f - (1.0f - (bb / 50.0f))) + 1.0f) + 1.0f) * (1.0f - (1.0f - (bb / 50.0f))) + 
                            (1.0f - (bb / 50.0f)) : softr);
        float softg = aa == 1.0f ? G : (G > aa ? (-1.0f / ((G - aa) / (bb - aa) + 1.0f) + 1.0f) * (bb - aa) + aa : G);
        float softG = bb == 1.0f ? softg : (softg > 1.0f - (bb / 50.0f) ? (-1.0f / ((softg - (1.0f - (bb / 50.0f))) / 
                            (1.0f - (1.0f - (bb / 50.0f))) + 1.0f) + 1.0f) * (1.0f - (1.0f - (bb / 50.0f))) + 
                            (1.0f - (bb / 50.0f)) : softg);
        float softb = aa == 1.0f ? B : (B > aa ? (-1.0f / ((B - aa) / (bb - aa) + 1.0f) + 1.0f) * (bb - aa) + aa : B);
        float softB = bb == 1.0f ? softb : (softb > 1.0f - (bb / 50.0f) ? (-1.0f / ((softb - (1.0f - (bb / 50.0f))) / 
                            (1.0f - (1.0f - (bb / 50.0f))) + 1.0f) + 1.0f) * (1.0f - (1.0f - (bb / 50.0f))) + 
                            (1.0f - (bb / 50.0f)) : softb);

        float Cr = (softR * -1.0f) + 1.0f;
        float Cg = (softG * -1.0f) + 1.0f;
        float Cb = (softB * -1.0f) + 1.0f;

        // Apply soft clipping final transform
        auto softClipFinal = [&](float C) -> float {
            float c = ss == 1.0f ? C : (C > ss ? (-1.0f / ((C - ss) / (sf - ss) + 1.0f) + 1.0f) * (sf - ss) + ss : C);
            return sf == 1.0f ? (c - 1.0f) * -1.0f : 
                                (((c > 1.0f - (-p_SoftClipF / 50.0f) ? (-1.0f / ((c - (1.0f - (-p_SoftClipF / 50.0f))) / 
                                (1.0f - (1.0f - (-p_SoftClipF / 50.0f))) + 1.0f) + 1.0f) * (1.0f - 
                                (1.0f - (-p_SoftClipF / 50.0f))) + (1.0f - (-p_SoftClipF / 50.0f)) : c) - 1.0f) * -1.0f);
        };

        float CR = softClipFinal(Cr);
        float CG = softClipFinal(Cg);
        float CB = softClipFinal(Cb);

        // Apply source specific transform
        auto sourceSpecificTransform = [&](float C) -> float {
            return p_Source == 0 ? C : 
                                  (C >= 0.0f && C <= 1.0f ? (C < 0.0181f ? (C * 4.5f) : 
                                  1.0993f * powf(C, 0.45f) - (1.0993f - 1.0f)) : C);
        };

        float SR = sourceSpecificTransform(CR);
        float SG = sourceSpecificTransform(CG);
        float SB = sourceSpecificTransform(CB);

        // Set output values with switch modifications
        auto setOutputWithSwitch = [&](float C) -> float {
            return p_SwitchA == 1 ? (C < 1.0f ? 1.0f : C) - 1.0f 
                                  : (p_SwitchB == 1 ? (C >= 0.0f ? 0.0f : C + 1.0f) : C);
        };

        p_Output[index] = setOutputWithSwitch(SR);
        p_Output[index + 1] = setOutputWithSwitch(SG);
        p_Output[index + 2] = setOutputWithSwitch(SB);
        p_Output[index + 3] = p_Input[index + 3];
    }
}