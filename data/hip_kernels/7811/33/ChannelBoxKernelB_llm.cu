#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ChannelBoxKernelB(const float* p_Input, float* p_Output, int p_Width, int p_Height, int p_Display) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < p_Width) && (y < p_Height)) {
        const int index = (y * p_Width + x) * 4;
        float alpha = p_Output[index + 3];

        if (p_Display == 1) {
            p_Output[index] = alpha;
            p_Output[index + 1] = alpha;
            p_Output[index + 2] = alpha;
        } else {
            p_Output[index] = p_Output[index] * alpha + p_Input[index] * (1.0f - alpha);
            p_Output[index + 1] = p_Output[index + 1] * alpha + p_Input[index + 1] * (1.0f - alpha);
            p_Output[index + 2] = p_Output[index + 2] * alpha + p_Input[index + 2] * (1.0f - alpha);
        }

        p_Output[index + 3] = 1.0f;
    }
}