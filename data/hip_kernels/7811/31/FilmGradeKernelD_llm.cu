#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FilmGradeKernelD(float* p_Input, float* p_Output, int p_Width, int p_Height, float p_Pivot, int p_Display) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Shared constants
    const float height = p_Height;
    const float width = p_Width;
    const float RES = width / 1920.0f;

    if (x < p_Width && y < p_Height) {
        const int index = (y * p_Width + x) * 4;
        const float X = x;
        const float Y = y;

        if (p_Display == 1) {
            p_Output[index] = (Y / height >= p_Pivot && Y / height <= p_Pivot + 0.005f * RES && fmodf(X, 2.0f) != 0.0f) ||
                              (p_Output[index] >= (Y - 5.0f * RES) / height && p_Output[index] <= (Y + 5.0f * RES) / height) ? 1.0f : 0.0f;
        } else if (p_Display == 2) {
            float overlay = (Y / height >= p_Pivot && Y / height <= p_Pivot + 0.005f * RES && fmodf(X, 2.0f) != 0.0f) ||
                            (p_Input[index] >= (Y - 5.0f * RES) / height && p_Input[index] <= (Y + 5.0f * RES) / height) ? 1.0f : 0.0f;
            p_Output[index] = overlay == 0.0f ? p_Output[index] : overlay;
        }
    }
}