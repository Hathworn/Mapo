#include "hip/hip_runtime.h"
```c++
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void k_Exposure(float* p_Input, int p_Width, int p_Height, float p_Exposure) {
    // Compute global thread indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread is within bounds
    if (x < p_Width && y < p_Height) {
        // Compute the linear index for 4-channel input
        const int index = (y * p_Width + x) * 4;

        // Precompute exposure factor
        float exposureFactor = exp2(p_Exposure);

        // Multiply each color channel by exposure factor
        p_Input[index] *= exposureFactor;
        p_Input[index + 1] *= exposureFactor;
        p_Input[index + 2] *= exposureFactor;
    }
}