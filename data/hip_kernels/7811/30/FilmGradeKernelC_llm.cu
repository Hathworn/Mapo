#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float Sat(float r, float g, float b) {
    float min = fminf(fminf(r, g), b);
    float max = fmaxf(fmaxf(r, g), b);
    float delta = max - min;
    float S = max != 0.0f ? delta / max : 0.0f;
    return S;
}

__global__ void FilmGradeKernelC(float* p_Input, int p_Width, int p_Height, float p_ContR, float p_ContG, float p_ContB, float p_SatR, float p_SatG, float p_SatB, float p_ContP) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < p_Width && y < p_Height) {
        const int index = (y * p_Width + x) * 4;
        float contR = (p_Input[index] - p_ContP) * p_ContR + p_ContP;
        float contG = (p_Input[index + 1] - p_ContP) * p_ContG + p_ContP;
        float contB = (p_Input[index + 2] - p_ContP) * p_ContB + p_ContP;

        // Compute luma using constants
        float luma = contR * 0.2126f + contG * 0.7152f + contB * 0.0722f;

        // Constants for saturation computation
        float satFactor = 1.0f - (p_SatR * 0.2126f + p_SatG * 0.7152f + p_SatB * 0.0722f);

        // Apply saturation to each channel
        float outR = satFactor * luma + contR * p_SatR;
        float outG = satFactor * luma + contG * p_SatG;
        float outB = satFactor * luma + contB * p_SatB;

        // Output the saturated values
        p_Input[index] = outR;
        p_Input[index + 1] = outG;
        p_Input[index + 2] = outB;
    }
}