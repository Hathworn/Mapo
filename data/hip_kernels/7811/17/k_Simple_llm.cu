#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void k_Simple(const float* p_Input, float* p_Output, int p_Width, int p_Height) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary check
    if (x < p_Width && y < p_Height) {
        const int index = (y * p_Width + x) * 4;

        // Use a loop to reduce redundant code
        for(int i = 0; i < 4; i++) {
            p_Output[index + i] = p_Input[index + i];
        }
    }
}