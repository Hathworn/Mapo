#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FilmGradeKernelA(float* p_Input, int p_Width, int p_Height, float p_Exp) {
    // Calculate global thread index
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Bound check to ensure that we are not accessing out of bounds memory
    if (x < p_Width && y < p_Height) {
        // Compute the linear index for the current pixel in the input array
        const int index = (y * p_Width + x) * 4;

        // Update the pixel value using the exponent parameter
        p_Input[index] += p_Exp * 0.01f;
    }
}