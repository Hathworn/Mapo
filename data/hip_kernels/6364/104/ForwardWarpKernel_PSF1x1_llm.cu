#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Calculate global indices for thread
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary check
    if (i >= h || j >= w) return;
    
    // Precompute offsets
    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Load flow vectors
    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Calculate target coordinates
    float cx = u_ * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v_ * time_scale + static_cast<float>(i) + 1.0f;

    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Load source pixel value
    float value = src[image_row_offset + j];

    // Ensure atomic update if within image bounds
    if ((tx >= 0) && (tx < w) && (ty >= 0) && (ty < h)) 
    {
        atomicAdd(dst + static_cast<size_t>(ty) * image_stride + tx, value);
    }
}