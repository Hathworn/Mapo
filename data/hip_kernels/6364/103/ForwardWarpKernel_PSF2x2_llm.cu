#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Compute bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    // Target pixel integer coords
    int tx = (int) px;
    int ty = (int) py;

    float value = src[image_row_offset + j];
    float weight;

    // Define a lambda function to reduce repetitive calculations
    auto accumulate = [&](int tx, int ty, float dx_weight, float dy_weight) {
        if (!(tx >= w || tx < 0 || ty >= h || ty < 0)) {
            weight = dx_weight * dy_weight;
            atomicAdd(dst + ty * image_stride + tx, value * weight);
            atomicAdd(normalization_factor + ty * image_stride + tx, weight);
        }
    };

    // Fill pixel containing bottom right corner
    accumulate(tx, ty, dx, dy);

    // Fill pixel containing bottom left corner
    accumulate(tx - 1, ty, 1.0f - dx, dy);

    // Fill pixel containing upper left corner
    accumulate(tx - 1, ty - 1, 1.0f - dx, 1.0f - dy);

    // Fill pixel containing upper right corner
    accumulate(tx, ty - 1, dx, 1.0f - dy);
}