#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid out-of-bounds memory access
    if (i < h && j < w) {
        const int pos = i * s + j;

        float scale = normalization_factor[pos];

        // Avoid division by zero
        float invScale = (scale != 0.0f) ? (1.0f / scale) : 1.0f;

        // Scale the image values
        image[pos] *= invScale;
    }
}