#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Calculate position and check bounds
    for (int pos = blockIdx.x * blockDim.x + threadIdx.x; pos < len; pos += gridDim.x * blockDim.x)
    {
        // Scale the vector
        d_res[pos] = d_src[pos] * scale;
    }
}