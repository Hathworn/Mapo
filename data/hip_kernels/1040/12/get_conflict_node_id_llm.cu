#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void get_conflict_node_id(int *deleted_rows, int *row_group, const int search_depth, int *conflict_node_id, const int total_dl_matrix_row_num) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < total_dl_matrix_row_num) { // Use global index calculation
        if (row_group[i] == search_depth + 1 && deleted_rows[i] < search_depth + 1) {
            atomicMax(conflict_node_id, deleted_rows[i]); // Atomic operation for updating
        }
    }
    __syncthreads(); // Synchronize threads within a block
}