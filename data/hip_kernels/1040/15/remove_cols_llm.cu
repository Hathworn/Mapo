#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void remove_cols(short *deleted_cols, int *col_group, const int conflict_col_id, const int total_dl_matrix_col_num) {
    for (int i = threadIdx.x; i < total_dl_matrix_col_num; i = i + blockDim.x) {
        if (col_group[i] == col_group[conflict_col_id]) {
            deleted_cols[i] = -1;
        }
    }
}

__global__ void remove_cols(short *deleted_cols, int *col_group, const int conflict_col_id, const int total_dl_matrix_col_num) {
    // Utilizing shared memory to minimize global memory access
    extern __shared__ int col_group_shared[];
    
    int tid = threadIdx.x;
    col_group_shared[tid] = col_group[tid];
    __syncthreads();

    for (int i = tid; i < total_dl_matrix_col_num; i += blockDim.x) {
        if (col_group_shared[i] == col_group_shared[conflict_col_id]) {
            deleted_cols[i] = -1;
        }
    }
}