#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void check_existance_of_candidate_rows(short *deleted_rows, int *row_group, const int search_depth, int *token, int *selected_row_id, const int total_dl_matrix_row_num) {
    for (int i = threadIdx.x; i < total_dl_matrix_row_num; i += blockDim.x) {
        if (deleted_rows[i] == 0 && row_group[i] == search_depth) {
            *token = 1;
            atomicMin(selected_row_id, i);
        }
    }
}

__global__ void check_existance_of_candidate_rows(int *deleted_rows, int *row_group, const int search_depth, int *token, int *selected_row_id, const int total_dl_matrix_row_num) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensuring threads work on valid indices
    if (idx < total_dl_matrix_row_num) {
        if (deleted_rows[idx] == 0 && row_group[idx] == search_depth) {
            atomicExch(token, 1);
            atomicMin(selected_row_id, idx);
        }
    }
}