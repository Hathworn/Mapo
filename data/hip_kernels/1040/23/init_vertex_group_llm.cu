#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void get_vertex_row_group(int *row_group, bool *dl_matrix, const int vertex_num, const int total_dl_matrix_row_num, const int total_dl_matrix_col_num) {
    int stride = blockDim.x * gridDim.x;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < total_dl_matrix_row_num; i += stride) {
        int row_sum = 0;
        for (int j = 0, cur_index = i * total_dl_matrix_col_num; j < vertex_num; j++, cur_index++) {
            row_sum += (int)(dl_matrix[cur_index]) * (j + 1);
        }
        row_group[i] = row_sum; // Use local variable and assign once for each row.
    }
}

__global__ void init_vertex_group(int *row_group, bool *dl_matrix, int *vertex_num, int *t_cn, int *t_rn, int *offset_row, int *offset_matrix, int graph_count) {
    int k = blockIdx.x;
    if (k < graph_count) {
        get_vertex_row_group(row_group + offset_row[k],
                             dl_matrix + offset_matrix[k], vertex_num[k], t_rn[k],
                             t_cn[k]);
    }
}