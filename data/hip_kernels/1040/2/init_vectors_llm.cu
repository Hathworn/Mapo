#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void init_vectors(short *vec, const int vec_length) {
    for (int i = threadIdx.x; i < vec_length; i += blockDim.x) {
        vec[i] = 0;
    }
}

__global__ void init_vectors(int *vec, const int vec_length) {
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique index
    int stride = blockDim.x * gridDim.x;               // Calculate the grid wide stride
    for (int i = index; i < vec_length; i += stride) { // Iterate over the array with grid stride loops
        vec[i] = 0;
    }
}