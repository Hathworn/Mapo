#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void add_gpu(int *device_var, int val) {
    atomicAdd(device_var, val);
}

__global__ void add_gpu(int *device_arr, int device_idx, int val) {
    // Using atomic operation to ensure safe concurrent updates
    atomicAdd(&(device_arr[device_idx]), val);
}