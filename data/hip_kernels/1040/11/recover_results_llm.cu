#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void recover_results(short *results, const int search_depth, const int total_dl_matrix_row_num) {
    for (int i = threadIdx.x; i < total_dl_matrix_row_num; i = i + blockDim.x) {
        if (results[i] == search_depth) {
            results[i] = 0;
        }
    }
}

__global__ void recover_results(int *results, const int search_depth, const int total_dl_matrix_row_num) {
    // Calculate global index for the current thread
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Iterate over data in steps of the total number of threads
    for (; idx < total_dl_matrix_row_num; idx += blockDim.x * gridDim.x) {
        if (results[idx] == search_depth) {
            results[idx] = 0;
        }
    }
}