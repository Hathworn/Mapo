#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void recover_deleted_rows(short *deleted_rows, const int search_depth, const int total_dl_matrix_row_num) {
    for (int i = threadIdx.x; i < total_dl_matrix_row_num; i += blockDim.x) {
        if (abs(deleted_rows[i]) >= search_depth) {
            deleted_rows[i] = 0;
        }
    }
}

__global__ void recover_deleted_rows(int *deleted_rows, const int search_depth, const int total_dl_matrix_row_num) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    for (int i = idx; i < total_dl_matrix_row_num; i += gridDim.x * blockDim.x) {
        if (abs(deleted_rows[i]) >= search_depth) {
            deleted_rows[i] = 0;
        }
    }
}