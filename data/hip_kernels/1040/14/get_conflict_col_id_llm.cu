#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void get_conflict_col_id(bool *dl_matrix, short *deleted_cols, int *conflict_col_id, int *conflict_edge, int total_dl_matrix_col_num, int vertex_num) {
    // Pointer to edge_a and edge_b in dl_matrix to reduce redundant calculations
    bool *edge_a_dlmatrix = dl_matrix + conflict_edge[0] * total_dl_matrix_col_num;
    bool *edge_b_dlmatrix = dl_matrix + conflict_edge[1] * total_dl_matrix_col_num;

    // Loop with step size as blockDim.x for conflict column checking
    for (int j = threadIdx.x; j < total_dl_matrix_col_num; j += blockDim.x) {
        // Check condition and use atomicMax for recording highest index
        if (edge_a_dlmatrix[j] == edge_b_dlmatrix[j] && deleted_cols[j] > 0 && edge_b_dlmatrix[j] == 1) {
            atomicMax(conflict_col_id, j);
        }
    }
}

__global__ void get_conflict_col_id(int *dl_matrix, int *deleted_cols, int *conflict_col_id, int *conflict_edge, int total_dl_matrix_col_num, int vertex_num) {
    // Calculate base indices to minimize repetitive calculations
    int base_a = conflict_edge[0] * total_dl_matrix_col_num;
    int base_b = conflict_edge[1] * total_dl_matrix_col_num;

    // Loop over columns with step size of blockDim.x
    for (int j = threadIdx.x; j < total_dl_matrix_col_num; j += blockDim.x) {
        // Use pre-computed base indices for value checks and atomicMax
        if (dl_matrix[base_a + j] == dl_matrix[base_b + j] && deleted_cols[j] > 0 && dl_matrix[base_b + j] == 1) {
            atomicMax(conflict_col_id, j);
        }
    }
    // Synchronize threads within block
    __syncthreads();
}