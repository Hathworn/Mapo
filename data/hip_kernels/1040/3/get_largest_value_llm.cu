#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void get_largest_value_device(short *vec, const int vec_length, int *max) {
    for (int i = threadIdx.x; i < vec_length; i += blockDim.x) {
        atomicMax(max, vec[i]);
    }
}

__global__ void get_largest_value(int *vec, const int vec_length, int* max) {
    // Use shared memory for intermediate maximums
    extern __shared__ int shared_max[];
    int local_max = INT_MIN;
    
    // Calculate local maximum
    for (int i = threadIdx.x; i < vec_length; i += blockDim.x) {
        local_max = max(local_max, vec[i]);
    }
    shared_max[threadIdx.x] = local_max;
    __syncthreads();
    
    // Find maximum in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            shared_max[threadIdx.x] = max(shared_max[threadIdx.x], shared_max[threadIdx.x + stride]);
        }
        __syncthreads();
    }
    
    // Use atomicMax to write the result to global memory
    if (threadIdx.x == 0) {
        atomicMax(max, shared_max[0]);
    }
}