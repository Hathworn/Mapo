#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void get_conflict_edge(bool *dl_matrix, short *deleted_rows, int *row_group, const int conflict_node_id, const int search_depth, int *conflict_edge, const int vertex_num, const int total_dl_matrix_row_num, const int total_dl_matrix_col_num) {

    for (int i = threadIdx.x; i < total_dl_matrix_row_num; i += blockDim.x) {
        // Check for negative conflict node id
        if (deleted_rows[i] == -conflict_node_id) {
            atomicMax(conflict_edge, i);
        }
        // Check for matching search depth and conflict node id
        if (row_group[i] == search_depth + 1 && deleted_rows[i] == conflict_node_id) {
            atomicMax(conflict_edge + 1, i);
        }
    }
}

__global__ void get_conflict_edge(int *dl_matrix, int *deleted_rows, int *deleted_cols, int *row_group, const int conflict_node_id, const int search_depth, int *conflict_edge, const int vertex_num, const int total_dl_matrix_row_num, const int total_dl_matrix_col_num) {

    for (int i = threadIdx.x; i < total_dl_matrix_row_num; i += blockDim.x) {
        // Check for negative conflict node id
        if (deleted_rows[i] == -conflict_node_id) {
            atomicMax(conflict_edge, i);
        }
        // Check for matching search depth and conflict node id
        if (row_group[i] == search_depth + 1 && deleted_rows[i] == conflict_node_id) {
            atomicMax(conflict_edge + 1, i);
        }
    }
    __syncthreads();
}