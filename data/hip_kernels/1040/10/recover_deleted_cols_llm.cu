#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void recover_deleted_cols(short *deleted_cols, const int search_depth, const int total_dl_matrix_col_num) {
    // Optimize loop to avoid repeated calculation of thread index range
    for (int i = threadIdx.x; i < total_dl_matrix_col_num; i += blockDim.x) {
        if (deleted_cols[i] >= search_depth) {
            deleted_cols[i] = 0;
        }
    }
}

__global__ void recover_deleted_cols(int *deleted_cols, const int search_depth, const int total_dl_matrix_col_num) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Reduce thread divergence by processing valid indices only
    for (int i = idx; i < total_dl_matrix_col_num; i += gridDim.x * blockDim.x) {
        if (deleted_cols[i] >= search_depth) {
            deleted_cols[i] = 0;
        }
    }
}