#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void find_index(short *vec, const int vec_length, int *value, int *index) {
    for (int i = threadIdx.x; i < vec_length; i += blockDim.x) {
        if (vec[i] == *value) {
            atomicMax(index, i);
        }
    }
}

__global__ void find_index(int *vec, const int vec_length, int *value, int *index) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID
    if (i < vec_length && vec[i] == *value) {      // Check bounds and value match
        atomicMax(index, i);
    }
}