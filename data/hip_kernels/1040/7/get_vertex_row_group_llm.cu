#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void get_vertex_row_group(int *row_group, bool *dl_matrix, const int vertex_num, const int total_dl_matrix_row_num, const int total_dl_matrix_col_num) {
    // Use thread synchronization for efficiency
    for (int i = threadIdx.x; i < total_dl_matrix_row_num; i += blockDim.x) {
        int sum = 0; // Reduce global memory access
        for (int j = 0, cur_index = i * total_dl_matrix_col_num; j < vertex_num; j++, cur_index++) {
            sum += (int)(dl_matrix[cur_index]) * (j + 1);
        }
        row_group[i] = sum;
    }
    __syncthreads(); // Ensure all threads complete before exit
}

__global__ void get_vertex_row_group(int *row_group, int *dl_matrix, const int vertex_num, const int total_dl_matrix_row_num, const int total_dl_matrix_col_num) {
    // Use optimized index calculation
    for (int i = threadIdx.x; i < total_dl_matrix_row_num; i += blockDim.x) {
        int sum = 0; // Accumulate results locally
        for (int j = 0; j < vertex_num; j++) {
            sum += dl_matrix[i * total_dl_matrix_col_num + j] * (j + 1);
        }
        row_group[i] = sum;
    }
    __syncthreads(); // Ensure memory consistency
}