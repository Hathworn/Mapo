#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
/*This file is part of quantumsim. (https://github.com/brianzi/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE.txt or https://www.gnu.org/licenses/gpl.txt*/

__global__ void two_qubit_ptm(double *dm, double *ptm_g, unsigned int bit0, unsigned int bit1, unsigned int no_qubits) {
    const unsigned int x = threadIdx.x;
    const unsigned int high_x = blockIdx.x * blockDim.x;

    extern __shared__ double ptm[];
    double *data = &ptm[256]; // Shared memory for data

    int higher_bit = max(bit0, bit1);
    int lower_bit = min(bit0, bit1);
    int high_mask = ~((1 << (2 * higher_bit + 2)) - 1);
    int mid_mask = (~((1 << (2 * lower_bit + 4)) - 1)) & (~high_mask);
    int low_mask = ~(high_mask | mid_mask) & (~0xf);

    int pos = high_x | x;
    int global_from =
        (pos & high_mask)
        | ((pos & mid_mask) >> 2)
        | ((pos & low_mask) >> 4)
        | ((pos & 0x3) << (2 * bit0))
        | (((pos & 0xc) >> 2) << (2 * bit1));

    // Load ptm to shared memory, ensuring all necessary values are loaded
    for (int i = x; i < 256; i += blockDim.x) {
        ptm[i] = ptm_g[i];
    }
    __syncthreads();

    if (high_x + x >= (1 << (2 * no_qubits))) return;

    // Load data block to shared memory
    data[x] = dm[global_from];
    __syncthreads();

    unsigned int row = x & 0xf;
    unsigned int idx = x & ~0xf;

    double acc = 0;
    // Accumulate the result using matrix-vector multiplication
    for (int i = 0; i < 16; i++) {
        acc += ptm[16 * row + i] * data[idx + i];
    }

    // Store the accumulated result
    dm[global_from] = acc;
}