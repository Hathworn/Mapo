#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*This file is part of quantumsim. (https://github.com/brianzi/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE.txt or https://www.gnu.org/licenses/gpl.txt*/

// kernel to transform to pauli basis (up, x, y, down)
// to be run on a complete complex density matrix, once for each bit
// this operation is its own inverse (can also be used in opposite direction)
__global__ void bit_to_pauli_basis(double *complex_dm, unsigned int mask, unsigned int no_qubits) {
    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    const double sqrt2 = 0.70710678118654752440;

    // Return if out of bounds
    if ((x >= (1 << no_qubits)) || (y >= (1 << no_qubits))) return;

    // Calculate addresses only once
    int x_masked = x & mask;
    int y_masked = y & mask;
    int x_unmasked = x & ~mask;
    int y_unmasked = y & ~mask;
    int b_addr = ((x_masked << 1) | x_unmasked) << no_qubits | y_unmasked;
    int c_addr = (x_unmasked << no_qubits) | ((y_masked << 1) | y_unmasked);
    
    double *b_ptr = complex_dm + (b_addr << 1);
    double *c_ptr = complex_dm + (c_addr << 1);

    if (x_masked && !y_masked) {
        double b = *b_ptr;
        double c = *c_ptr;
        *b_ptr = (b + c) * sqrt2;
        *c_ptr = (b - c) * sqrt2;
    }

    if (!x_masked && y_masked) {
        b_ptr++;
        c_ptr++;
        double b = *b_ptr;
        double c = *c_ptr;
        *b_ptr = (b + c) * sqrt2;
        *c_ptr = (b - c) * sqrt2;
    }
}