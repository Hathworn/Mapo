#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*This file is part of quantumsim. (https://github.com/brianzi/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE.txt or https://www.gnu.org/licenses/gpl.txt*/


//kernel to transform to pauli basis (up, x, y, down)
//to be run on a complete complex density matrix, once for each bit
//this operation is its own inverse (can also be used in opposite direction)
__global__ void swap(double *dm, unsigned int bit1, unsigned int bit2, unsigned int no_qubits) {
    unsigned int addr = threadIdx.x + blockDim.x * blockIdx.x;

    if (addr >= (1 << (2 * no_qubits))) return;

    unsigned int bit1_mask = (0x3 << (2 * bit1));
    unsigned int bit2_mask = (0x3 << (2 * bit2));

    // Calculate addr2 without separate masking and shifting
    unsigned int addr2 = addr ^ (((addr >> (2 * bit1)) ^ (addr >> (2 * bit2))) & 0x3) << (2 * bit1) 
                                ^ (((addr >> (2 * bit1)) ^ (addr >> (2 * bit2))) & 0x3) << (2 * bit2);

    // Perform swap if necessary
    if (addr < addr2) {
        double t = dm[addr];
        dm[addr] = dm[addr2];
        dm[addr2] = t;
    }
}