#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*This file is part of quantumsim. (https://github.com/brianzi/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE.txt or https://www.gnu.org/licenses/gpl.txt*/

//kernel to transform to pauli basis (up, x, y, down)
//to be run on a complete complex density matrix, once for each bit
//this operation is its own inverse (can also be used in opposite direction)
__global__ void get_diag(double *dm9, double *out, unsigned int no_qubits) {
    // Calculate linear global thread index
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Ensure we do not operate on indices beyond the data array
    if (x >= (1 << no_qubits)) return;

    // Precompute shifted index from original, assuming bits should interleave
    unsigned int addr_real = 0;
    unsigned int mask = 1U;
    for (int i = 0; i < 16; i++) {
        // Alternate shifting
        addr_real |= ((x & mask) << i) | ((x & mask) << (i + 1));
        mask <<= 1;
    }
    out[x] = dm9[addr_real];
}