#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*This file is part of quantumsim. (https://github.com/brianzi/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE.txt or https://www.gnu.org/licenses/gpl.txt*/

__global__ void trace(double *diag, int bit) {
    unsigned int x = threadIdx.x;
    unsigned int mask = (bit >= 0) ? (1 << bit) : 0;

    extern __shared__ double s_diag[];

    // Load data into shared memory
    s_diag[x] = diag[x];
    __syncthreads();

    double a = 0.0;

    // Use shared memory to avoid redundant memory access
    for (unsigned int i = 1; i < blockDim.x; i <<= 1) {
        if (i != mask && x >= i) {
            a = s_diag[x - i];
        }
        __syncthreads();
        if (i != mask && x >= i) {
            s_diag[x] += a;
        }
        __syncthreads();
    }

    // Write to global memory conditionally
    if (x == 0) {
        diag[blockIdx.x] = s_diag[blockDim.x - 1];
    } else if (x == 1 && bit >= 0) {
        diag[blockIdx.x + 1] = s_diag[blockDim.x - 1 - mask];
    }
}