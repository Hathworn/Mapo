#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to transform to Pauli basis (up, x, y, down)
// This operation is its own inverse and can be used in the opposite direction
__global__ void dm_reduce(double *dm, unsigned int bit, double *dm0, unsigned int state, unsigned int no_qubits) {

    // Calculate the global thread address
    const int addr = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Terminate threads that exceed matrix size
    if(addr >= (1 << (2 * no_qubits))) return;
    
    // Masks to extract bits related to the specified qubit
    const int low_mask = (1 << (2 * bit)) - 1;  // Mask for lower bits
    const int high_mask = (~low_mask) << 2;     // Mask for higher bits
    
    // Check if the relevant bits match the specified state
    if(((addr >> (2 * bit)) & 0x3) == state) {
        // Compute reduced address and perform transformation
        dm0[(addr & low_mask) | ((addr & high_mask) >> 2)] = dm[addr];
    }
}