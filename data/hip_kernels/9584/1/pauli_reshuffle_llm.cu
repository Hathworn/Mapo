#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*This file is part of quantumsim. (https://github.com/brianzi/quantumsim)*/
/*(c) 2016 Brian Tarasinski*/
/*Distributed under the GNU GPLv3. See LICENSE.txt or https://www.gnu.org/licenses/gpl.txt*/

//kernel to transform to pauli basis (up, x, y, down)
//to be run on a complete complex density matrix, once for each bit
//this operation is its own inverse (can also be used in opposite direction)
__global__ void pauli_reshuffle(double *complex_dm, double *real_dm, unsigned int no_qubits, unsigned int direction) {

    const int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    const int size = 1 << no_qubits;
    
    // Early exit for threads out of bounds
    if (x >= size || y >= size) return;

    // Calculate if imaginary part is needed
    unsigned int v = ~x & y;
    unsigned int py = __popc(v) & 0x3;  // Calculate the population count using intrinsic

    // Calculate complex address
    const unsigned int addr_complex = (((x << no_qubits) | y) << 1) + (py & 1);

    // Interleave address for pauli basis
    unsigned int addr_real = 0;
    for (int i = 0; i < no_qubits; i++) {  // Use `no_qubits` instead of hardcoded value
        addr_real |= ((x & (1U << i)) << i) | ((y & (1U << i)) << (i + 1));
    }

    double sign = (py == 3 || py == 2) ? -1.0 : 1.0;

    // Avoid branch divergence
    if (direction == 0) {
        real_dm[addr_real] = sign * complex_dm[addr_complex];
    } else {
        complex_dm[addr_complex] = sign * real_dm[addr_real];
    }
}