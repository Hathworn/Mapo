#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void cal_pi(float *sum, int nbin, float step, int nthreads, int nblocks)
{
    float x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Sequential thread index

    float local_sum = 0.0f; // Use local variable for reduction to minimize global memory access
    for (int i = idx; i < nbin; i += nthreads * nblocks) {
        x = (i + 0.5f) * step;
        local_sum += 4.0f / (1.0f + x * x);
    }
    sum[idx] = local_sum; // Update global memory once per thread
}