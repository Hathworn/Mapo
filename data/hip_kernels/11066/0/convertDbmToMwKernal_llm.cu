#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// GPU function to converts the provided dBm value to mW.
// The power in milliwatts (P(mW)) is equal to 1mW times 10 raised by the
// power in decibel-milliwatts (P(dBm)) divided by 10:
// P(mW) = 1mW * 10 ^ (P(dBm) / 10)
__device__ float convertDbmToMw(const float dBm)
{
    return powf(10.0f, dBm / 10.0f);
}

__global__ void convertDbmToMwKernal(float *result, const float *data, unsigned int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (i < size) {
        // Compute dBm to mW conversion for valid index
        result[i] = convertDbmToMw(data[i]);
    }
}
```
