#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Perform division in parallel to handle large data efficiently
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {  // Only one thread performs the division
        *accuracy /= N;
    }
}