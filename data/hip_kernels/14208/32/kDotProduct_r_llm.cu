#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DP_BLOCKSIZE 512

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];
    
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;
    
    // Load elements into shared memory and calculate partial dot products
    if (eidx < numCols) {
        for (; eidx < numElements; eidx += numCols) {
            sum += a[eidx] * b[eidx];
        }
    }
    
    shmem[threadIdx.x] = sum;
    __syncthreads();
    
    // Perform reduction using binary tree method within warp to reduce total memory accesses
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            shmem[threadIdx.x] += shmem[threadIdx.x + i];
        }
        __syncthreads();
    }
    
    // Write the result of this block's dot product reduction to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}