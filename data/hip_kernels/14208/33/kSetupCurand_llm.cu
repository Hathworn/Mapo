#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate thread id using block-level abstractions
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    // Initialize hiprand state with distinct sequence
    hiprand_init(seed, tidx, 0, &state[tidx]);
}