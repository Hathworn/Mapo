#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceGmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 4;

    // unrolling 4 with boundary checks
    if (idx < n)
    {
        int sum = 0;
        if (idx < n) sum += g_idata[idx];
        if (idx + blockDim.x < n) sum += g_idata[idx + blockDim.x];
        if (idx + 2 * blockDim.x < n) sum += g_idata[idx + 2 * blockDim.x];
        if (idx + 3 * blockDim.x < n) sum += g_idata[idx + 3 * blockDim.x];
        g_idata[idx] = sum;
    }

    __syncthreads();

    // In-place reduction using loop for better readability
    for(int offset = blockDim.x / 2; offset > 32; offset /= 2) 
    {
        if (tid < offset)
            idata[tid] += idata[tid + offset];
        __syncthreads();
    }

    // Unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}