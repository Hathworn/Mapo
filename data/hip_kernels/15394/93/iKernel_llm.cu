#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void iKernel(float *src, float *dst, int n)
{
    // Calculate global index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds to prevent out-of-range memory access
    if (idx < n)
    {
        dst[idx] = src[idx] * 2.0f;
    }
}