#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lots_of_double_compute(double *inputs, int N, size_t niters, double *outputs)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t nthreads = gridDim.x * blockDim.x;

    for (; tid < N; tid += nthreads)
    {
        size_t iter;
        double val = inputs[tid];

        // Consolidate calculations for reduced operations within loop
        double offset1 = (5.0 - 101.0) / 3.0 + 102.0;
        double offset2 = (1.07 - 103.0) / 1.037 + 104.0;
        double offset3 = (3.00 - 105.0) / 0.22 + 106.0;

        for (iter = 0; iter < niters; iter++)
        {
            val = ((val + offset1) + offset2) + offset3;
        }

        outputs[tid] = val;
    }
}