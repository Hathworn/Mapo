#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffsetUnroll4(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate the starting index for this thread, considering unrolling
    unsigned int i = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    unsigned int k = i + offset;

    // Load data in registers for maximizing memory throughput
    float a_val1, b_val1, a_val2, b_val2, a_val3, b_val3, a_val4, b_val4;
    
    // Check if reads are within bounds and process in batches
    if (k < n) {
        a_val1 = A[k]; 
        b_val1 = B[k];
        C[i] = a_val1 + b_val1;
    }
    if (k + blockDim.x < n) {
        a_val2 = A[k + blockDim.x]; 
        b_val2 = B[k + blockDim.x];
        C[i + blockDim.x] = a_val2 + b_val2;
    }
    if (k + 2 * blockDim.x < n) {
        a_val3 = A[k + 2 * blockDim.x]; 
        b_val3 = B[k + 2 * blockDim.x];
        C[i + 2 * blockDim.x] = a_val3 + b_val3;
    }
    if (k + 3 * blockDim.x < n) {
        a_val4 = A[k + 3 * blockDim.x]; 
        b_val4 = B[k + 3 * blockDim.x];
        C[i + 3 * blockDim.x] = a_val4 + b_val4;
    }
}