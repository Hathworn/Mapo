#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unsafe(int *shared_var, int *values_read, int N, int iters)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N) return;

    // Load shared variable once
    int local_var = atomicAdd(shared_var, 1);
    values_read[tid] = local_var;

    // Use loop-specific increments in shared memory updates
    for (int i = 0; i < iters; i++)
    {
        atomicAdd(shared_var, 1);
    }
}