#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmem(int *g_idata, int *g_odata, unsigned int n)
{
    // Use dynamically allocated shared memory for flexible block size
    extern __shared__ int smem[];

    // Set thread ID
    unsigned int tid = threadIdx.x;

    // Boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Load data into shared memory
    smem[tid] = idata[tid];
    __syncthreads();

    // Reduce shared memory using loop unrolling for better performance
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            smem[tid] += smem[tid + s];
        }
        __syncthreads(); // Sync at each step
    }

    // Unrolled reduction for last warp
    if (tid < 32) {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}