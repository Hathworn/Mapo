#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved global kernel function using coalesced memory access
__global__ void copyCol(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure index is within bounds
    if (ix < nx && iy < ny)
    {
        // Use coalesced access pattern by swapping loop order
        out[iy * nx + ix] = in[iy * nx + ix];
    }
}