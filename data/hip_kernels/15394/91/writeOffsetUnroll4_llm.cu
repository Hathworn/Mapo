#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeOffsetUnroll4(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Use loop unrolling to minimize loop overhead and increase instruction-level parallelism
    #pragma unroll
    for (int j = 0; j < 4; j++)
    {
        int idx = k + j * blockDim.x;
        if (idx < n)
        {
            C[idx] = A[i + j * blockDim.x] + B[i + j * blockDim.x];
        }
    }
}