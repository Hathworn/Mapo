#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate unique thread index in the grid
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y; // Change to 2D indexing

    if (ix < nx && iy < ny) {
        unsigned int idx = iy * nx + ix; // Use ix and iy for indexing
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}