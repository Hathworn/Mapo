#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refill_randoms(float *dRand, int N, hiprandState *states)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;
    hiprandState state = states[tid]; // Load state for tid

    // Unroll loop for better performance
    for (int i = tid; i < N; i += nthreads * 4)
    {
        if (i < N) dRand[i] = hiprand_uniform(&state);
        if (i + nthreads < N) dRand[i + nthreads] = hiprand_uniform(&state);
        if (i + 2 * nthreads < N) dRand[i + 2 * nthreads] = hiprand_uniform(&state);
        if (i + 3 * nthreads < N) dRand[i + 3 * nthreads] = hiprand_uniform(&state);
    }

    states[tid] = state; // Save the updated state back
}