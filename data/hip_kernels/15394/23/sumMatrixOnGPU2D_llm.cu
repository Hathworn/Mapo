#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    // Use shared memory for faster access
    __shared__ float tileA[32][32];
    __shared__ float tileB[32][32];

    if (ix < nx && iy < ny) {
        unsigned int localIdxX = threadIdx.x;
        unsigned int localIdxY = threadIdx.y;
      
        tileA[localIdxY][localIdxX] = MatA[iy * nx + ix];
        tileB[localIdxY][localIdxX] = MatB[iy * nx + ix];

        __syncthreads(); // Ensure all threads have copied to shared memory

        MatC[iy * nx + ix] = tileA[localIdxY][localIdxX] + tileB[localIdxY][localIdxX];
    }
}