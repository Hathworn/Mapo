#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceGmem(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // Optimize in-place reduction using unrolling
    if (blockDim.x >= 1024)
    {
        if (tid < 512) idata[tid] += idata[tid + 512];
        __syncthreads();
    }
    
    if (blockDim.x >= 512)
    {
        if (tid < 256) idata[tid] += idata[tid + 256];
        __syncthreads();
    }
    
    if (blockDim.x >= 256)
    {
        if (tid < 128) idata[tid] += idata[tid + 128];
        __syncthreads();
    }
    
    if (blockDim.x >= 128)
    {
        if (tid < 64) idata[tid] += idata[tid + 64];
        __syncthreads();
    }

    // Unrolling warp without volatile (combined stages)
    if (tid < 32)
    {
        idata[tid] += idata[tid + 32];
        idata[tid] += idata[tid + 16];
        idata[tid] += idata[tid + 8];
        idata[tid] += idata[tid + 4];
        idata[tid] += idata[tid + 2];
        idata[tid] += idata[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}