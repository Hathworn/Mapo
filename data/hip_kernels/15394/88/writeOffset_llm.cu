#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global thread index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the offset index
    unsigned int k = i + offset;

    // Check bounds before writing to C
    if (k < n) {
        // Perform vector addition
        C[k] = A[i] + B[i];
    }
}