#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Process only if idx is within the range
    if (idx < N)
    {
        // Remove unnecessary loop
        C[idx] = A[idx] + B[idx];
    }
}