#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use warp divergence to prevent unnecessary calculations
    #pragma unroll
    for (int j = i; j < N; j += blockDim.x * gridDim.x)
    {
        C[j] = A[j] + B[j];
    }
}