#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *g_data, float value)
{
    // Use shared memory for faster access within a block
    __shared__ float s_value;
    
    // Load value to shared memory outside the loop
    if (threadIdx.x == 0)
    {
        s_value = value;
    }
    __syncthreads();
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform the addition using shared memory
    g_data[idx] += s_value;
}