#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use more threads per block for better occupancy if necessary
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for grid-stride loop

    // Use grid-stride loop for better workload distribution
    for(; i < N; i += stride) {
        C[i] = A[i] + B[i];
    }
}