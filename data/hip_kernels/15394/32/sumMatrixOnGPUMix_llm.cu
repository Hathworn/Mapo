#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny) {
    // Calculate 2D block and grid indices
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (ix < nx && iy < ny) {
        // Calculate 1D global index for matrix
        unsigned int idx = iy * nx + ix;
        // Perform element-wise matrix addition
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}