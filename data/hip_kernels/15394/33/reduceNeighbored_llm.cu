#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighbored (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if (idx >= n) return;

    // In-place reduction in shared memory to reduce global memory accesses
    extern __shared__ int sdata[];
    
    sdata[tid] = idata[tid];
    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if (tid % (2 * stride) == 0)
        {
            sdata[tid] += sdata[tid + stride];
        }
        // Synchronize within threadblock
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}