#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate 1D flattened index from 2D grid and block
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;

    // Check boundary conditions and perform addition
    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}