#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighbored (int *g_idata, int *g_odata, unsigned int n) {
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;

    // Boundary check
    if (idx >= n) return;

    // Using shared memory to reduce global memory accesses
    extern __shared__ int sdata[];
    sdata[tid] = g_idata[idx];
    __syncthreads();

    // In-place reduction in shared memory
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        if (tid % (2 * stride) == 0) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads(); // Synchronize within threadblock
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}