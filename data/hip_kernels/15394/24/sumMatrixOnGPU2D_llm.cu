#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate unique global thread ID
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure threads are within matrix bounds
    if (ix < nx && iy < ny) 
    {
        unsigned int idx = iy * nx + ix; // Compute linear index only for valid threads
        MatC[idx] = MatA[idx] + MatB[idx]; // Perform element-wise addition
    }
}