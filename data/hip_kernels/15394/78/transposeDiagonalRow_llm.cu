#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalRow(float *out, float *in, const int nx, const int ny)
{
    // Calculate linear thread index for grid stride loop
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int gridSize = blockDim.x * gridDim.x;
    
    // Optimize memory access pattern with grid stride loop
    for (unsigned int index = tid; index < nx * ny; index += gridSize) {
        unsigned int ix = index / ny; // Calculate x coordinate
        unsigned int iy = index % ny; // Calculate y coordinate

        if (ix < nx && iy < ny) {
            out[ix * ny + iy] = in[iy * nx + ix];
        }
    }
}