#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *F, double *D)
{
    // Calculate the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Only one thread is responsible for updating *F and *D
    if (tid == 0)
    {
        *F = __float2half_rn(12.1f); // Use half-precision conversion
        *D = 12.1;
    }
}