#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeUnroll4Row(float *out, float *in, const int nx, const int ny)
{
    // Calculate the starting point index for the row and column
    unsigned int ix = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds for the matrix dimensions
    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        unsigned int ti = iy * nx + ix; // access input in rows
        unsigned int to = ix * ny + iy; // access output in columns
        
        // Perform unrolled transpose operation
        float val1 = in[ti]; 
        float val2 = in[ti + blockDim.x];
        float val3 = in[ti + 2 * blockDim.x];
        float val4 = in[ti + 3 * blockDim.x];
        
        out[to] = val1;
        out[to + ny * blockDim.x] = val2;
        out[to + ny * 2 * blockDim.x] = val3;
        out[to + ny * 3 * blockDim.x] = val4;
    }
}