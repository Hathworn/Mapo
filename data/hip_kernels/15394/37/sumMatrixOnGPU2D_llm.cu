#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, int NX, int NY)
{
    // Calculate 1D index based on local thread ID and ensure within bounds
    unsigned int idx = blockIdx.y * blockDim.y * NX + blockIdx.x * blockDim.x + threadIdx.y * NX + threadIdx.x;
    
    if (threadIdx.x + blockDim.x * blockIdx.x < NX && threadIdx.y + blockDim.y * blockIdx.y < NY)
    {
        C[idx] = A[idx] + B[idx];
    }
}