#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffsetUnroll2(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int k = i + offset;

    // Check bounds and calculate in one statement where possible
    if (k < n) {
        C[i] = A[k] + B[k];
    }
    k += blockDim.x; // Compute k + blockDim.x in advance to optimize the memory access pattern
    if (k < n) {
        C[i + blockDim.x] = A[k] + B[k];
    }
}