#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeDiagonalCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate unique block index
    unsigned int blk_x = blockIdx.x;
    unsigned int blk_y = blockIdx.y;

    // Calculate global thread indices
    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    // Ensure within bounds
    if (ix < nx && iy < ny)
    {
        // Perform the transpose
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}