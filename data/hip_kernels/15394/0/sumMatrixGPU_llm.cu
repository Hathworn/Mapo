#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate thread's global index
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;

    // Check bounds and perform operation
    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}