#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighboredLess(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if (idx >= n) return;

    // Improved in-place reduction in shared memory for better memory access patterns
    __shared__ int smem[1024]; // Assuming blockDim.x <= 1024
    smem[tid] = idata[tid];
    __syncthreads();

    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        int index = 2 * stride * tid;
        if (index < blockDim.x)
        {
            smem[index] += smem[index + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}