#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refill_randoms(float *dRand, int N, hiprandState *states)
{
    // Calculate unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;
    
    // Use local state to optimize memory accesses
    hiprandState localState = states[tid];

    for (int i = tid; i < N; i += nthreads)
    {
        // Generate random number using local state
        dRand[i] = hiprand_uniform(&localState);
    }
    
    // Update global state with local state
    states[tid] = localState;
}