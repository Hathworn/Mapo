#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void checkIndex(void)
{
    // Cache commonly accessed thread and block indices
    int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z;
    int bx = blockIdx.x, by = blockIdx.y, bz = blockIdx.z;
    int bdx = blockDim.x, bdy = blockDim.y, bdz = blockDim.z;
    int gdx = gridDim.x, gdy = gridDim.y, gdz = gridDim.z;

    printf("threadIdx:(%d, %d, %d)\n", tx, ty, tz);
    printf("blockIdx:(%d, %d, %d)\n", bx, by, bz);

    printf("blockDim:(%d, %d, %d)\n", bdx, bdy, bdz);
    printf("gridDim:(%d, %d, %d)\n", gdx, gdy, gdz);
}