#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(float *out, float *in, const int nx, const int ny)
{
    // Calculate the linear index directly for improved memory access
    unsigned int idx = blockIdx.y * blockDim.y * nx + blockIdx.x * blockDim.x + threadIdx.y * nx + threadIdx.x;

    if (threadIdx.x + blockIdx.x * blockDim.x < nx && threadIdx.y + blockIdx.y * blockDim.y < ny)
    {
        out[idx] = in[idx];
    }
}