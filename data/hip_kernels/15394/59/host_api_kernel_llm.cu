#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void host_api_kernel(float *randomValues, float *out, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;

    // Utilize a stride loop for improved memory access and coalescing
    for (int i = tid; i < N; i += nthreads)
    {
        out[i] = randomValues[i] * 2;  // Inline operation to avoid temporary variable
    }
}