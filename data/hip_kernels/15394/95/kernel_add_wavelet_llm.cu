#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_add_wavelet(float *g_u2, float wavelets, const int nx, const int ny, const int ngpus) 
{
    // Global grid index calculation for y dimension
    int ipos = (ngpus == 2 ? ny - 10 : ny / 2 - 10);

    // Global x index for current thread
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;

    // Only one thread executes if condition, reducing warp divergence
    if (ix == nx / 2) 
    {
        unsigned int idx = ipos * nx + ix;  // Compute its index
        g_u2[idx] += wavelets;  // Add wavelet
    }
}