#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNaiveCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate global thread index in x and y directions
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Ensure within bounds to avoid unnecessary execution and out of bounds access
    if (ix < nx && iy < ny)
    {
        // Efficient coalesced global memory access pattern
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}