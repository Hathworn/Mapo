#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighboredGmem(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // optimize: loop unrolling for efficiency
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        int index = 2 * stride * tid;
        if (index < blockDim.x)
        {
            idata[index] += idata[index + stride];
        }
        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}