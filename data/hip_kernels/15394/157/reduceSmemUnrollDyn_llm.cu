#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceSmemUnrollDyn(int *g_idata, int *g_odata, unsigned int n)
{
    extern __shared__ int smem[];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // unrolling 4
    int tmpSum = 0;

    // Load data and sum: coalesced access, loop unrolling
    if (idx < n) {
        int a1 = g_idata[idx];
        int a2 = (idx + blockDim.x < n) ? g_idata[idx + blockDim.x] : 0;
        int a3 = (idx + 2 * blockDim.x < n) ? g_idata[idx + 2 * blockDim.x] : 0;
        int a4 = (idx + 3 * blockDim.x < n) ? g_idata[idx + 3 * blockDim.x] : 0;
        tmpSum = a1 + a2 + a3 + a4;
    }

    // store local sum into shared memory
    smem[tid] = tmpSum;
    __syncthreads();

    // in-place reduction in shared memory, handling full warp
    if (blockDim.x >= 1024) { if (tid < 512) smem[tid] += smem[tid + 512]; __syncthreads(); }
    if (blockDim.x >= 512) { if (tid < 256) smem[tid] += smem[tid + 256]; __syncthreads(); }
    if (blockDim.x >= 256) { if (tid < 128) smem[tid] += smem[tid + 128]; __syncthreads(); }
    if (blockDim.x >= 128) { if (tid < 64) smem[tid] += smem[tid + 64]; __syncthreads(); }

    // Unrolling warp-based reduction
    if (tid < 32) {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}