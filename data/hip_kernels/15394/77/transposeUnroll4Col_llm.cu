#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeUnroll4Col(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    // Use shared memory for coalesced memory access
    __shared__ float tile[4 * blockDim.x];

    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        tile[threadIdx.x] = in[to];
        tile[threadIdx.x + blockDim.x] = in[to + blockDim.x * ny];
        tile[threadIdx.x + 2 * blockDim.x] = in[to + 2 * blockDim.x * ny];
        tile[threadIdx.x + 3 * blockDim.x] = in[to + 3 * blockDim.x * ny];
    }
    __syncthreads();

    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        out[ti]                = tile[threadIdx.x];
        out[ti +   blockDim.x] = tile[threadIdx.x +   blockDim.x];
        out[ti + 2 * blockDim.x] = tile[threadIdx.x + 2 * blockDim.x];
        out[ti + 3 * blockDim.x] = tile[threadIdx.x + 3 * blockDim.x];
    }
}