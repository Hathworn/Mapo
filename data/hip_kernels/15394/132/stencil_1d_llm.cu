#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BDIM 256
#define RADIUS 3
__constant__ float coef[RADIUS + 1];

__global__ void stencil_1d(float *in, float *out, int N) {
    // Shared memory
    __shared__ float smem[BDIM + 2 * RADIUS];

    // Index to global memory
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop over the grid until all elements are processed
    while (idx < N) {
        // Index to shared memory for stencil calculation
        int sidx = threadIdx.x + RADIUS;

        // Read data from global memory into shared memory
        smem[sidx] = in[idx];

        // Read halo part to shared memory
        if (threadIdx.x < RADIUS) {
            // Avoids out-of-bound memory reads
            if (idx >= RADIUS) {
                smem[sidx - RADIUS] = in[idx - RADIUS];
            }
            if (idx + BDIM < N) {
                smem[sidx + BDIM] = in[idx + BDIM];
            }
        }

        // Synchronize (ensure all the data is available)
        __syncthreads();

        // Apply the stencil
        float tmp = 0.0f;

        #pragma unroll
        for (int i = 1; i <= RADIUS; i++) {
            tmp += coef[i] * (smem[sidx + i] - smem[sidx - i]);
        }

        // Store the result
        if (idx < N) {
            out[idx] = tmp;
        }

        // Update idx for next iteration
        idx += gridDim.x * blockDim.x;
    }
}