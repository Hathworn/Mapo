#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate global thread ID
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    // Check if thread is within matrix bounds and compute the result
    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}