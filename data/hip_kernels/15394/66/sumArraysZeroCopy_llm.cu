#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysZeroCopy(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use warp-synchronous programming to coalesce memory accesses
    if (i < N) {
        float a = A[i];
        float b = B[i];
        C[i] = a + b;
    }
}