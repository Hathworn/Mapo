#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeOptimizedRow(float *out, float *in, const int nx, const int ny)
{
    // Use shared memory to optimize global memory access
    __shared__ float tile[32][33]; // Avoid bank conflicts with padding
    
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    
    if (ix < nx && iy < ny)
    {
        // Load from global memory to shared memory
        tile[threadIdx.y][threadIdx.x] = in[iy * nx + ix];
    }
    __syncthreads();
    
    // Calculate transposed indices
    unsigned int transposed_ix = blockDim.y * blockIdx.y + threadIdx.x;
    unsigned int transposed_iy = blockDim.x * blockIdx.x + threadIdx.y;

    if (transposed_ix < ny && transposed_iy < nx)
    {
        // Write from shared memory to global memory
        out[transposed_iy * ny + transposed_ix] = tile[threadIdx.x][threadIdx.y];
    }
}