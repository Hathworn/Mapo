#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyRow(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Combine the index calculation to reduce repeated computation
    unsigned int index = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        out[index] = in[index];
    }
}