#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_2dfd_last(float *g_u1, float *g_u2, const int nx, const int iStart, const int iEnd)
{
    // Global index
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idx = ix + iStart * nx;
    
    // Shared memory index
    unsigned int stx = threadIdx.x + NPAD;
    
    // Shared memory for u2
    __shared__ float tile[BDIMX + NPAD2];
    
    const float alpha = 0.12f;
    
    // Register for y values
    float yval[9];

    // Preload values into yval
    for (int i = 0; i < 8; i++) yval[i] = g_u2[idx + (i - 4) * nx];
    int iskip = NPAD * nx;

    // Iterate over y dimension
    for (int iy = iStart; iy < iEnd; iy++)
    {
        yval[8] = g_u2[idx + iskip];

        // Load halo and main elements into shared memory
        if (threadIdx.x < NPAD)
        {
            tile[threadIdx.x] = g_u2[idx - NPAD];
            tile[stx + BDIMX] = g_u2[idx + BDIMX];
        }
        
        tile[stx] = yval[4];
        __syncthreads();

        if ((ix >= NPAD) && (ix < nx - NPAD))
        {
            float tmp = coef[0] * tile[stx] * 2.0f;

            // Compute stencil in x direction
            #pragma unroll
            for (int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (tile[stx - d] + tile[stx + d]);
            }

            // Compute stencil in y direction
            #pragma unroll
            for (int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (yval[4 - d] + yval[4 + d]);
            }

            // Time update
            g_u1[idx] = yval[4] + yval[4] - g_u1[idx] + alpha * tmp;
        }

        // Shift down yval for the next iteration
        #pragma unroll 8
        for (int i = 0; i < 8; i++)
        {
            yval[i] = yval[i + 1];
        }

        // Advance global index for next y plane
        idx += nx;
        __syncthreads();
    }
}