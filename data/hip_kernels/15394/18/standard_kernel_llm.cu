#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void standard_kernel(float a, float *out, int iters)
{
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

    if(tid == 0)
    {
        // Calculate powf once outside the loop since it's independent of i
        float tmp = powf(a, 2.0f);

        // Replace loop with a single assignment as all iterations do the same task
        *out = tmp;
    }
}