#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Use shared memory to optimize memory accesses
    extern __shared__ float sharedMem[];
    float* sharedMatA = sharedMem;
    float* sharedMatB = sharedMem + blockDim.x * ny;

    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;

    if (ix < nx) {
        for (int iy = 0; iy < ny; iy++) {
            int idx = iy * nx + ix;
            
            // Load data to shared memory
            sharedMatA[threadIdx.x * ny + iy] = MatA[idx];
            sharedMatB[threadIdx.x * ny + iy] = MatB[idx];

            // Synchronize to make sure all data is loaded
            __syncthreads();

            // Perform computation using shared memory
            MatC[idx] = sharedMatA[threadIdx.x * ny + iy] + sharedMatB[threadIdx.x * ny + iy];

            // Synchronize again before the next iteration
            __syncthreads();
        }
    }
}