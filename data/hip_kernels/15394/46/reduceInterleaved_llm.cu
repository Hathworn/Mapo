#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceInterleaved(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // in-place reduction in shared memory for faster access
    __shared__ int sdata[1024]; // assuming max blockDim.x = 1024
    sdata[tid] = idata[tid];
    __syncthreads();

    // Interleaved reduction using shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) 
    {
        if (tid < stride)
        {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}