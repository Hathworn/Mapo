#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FULL_MASK 0xffffffff

__global__ void reduceSmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // static shared memory
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // global index
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // unrolling 4 blocks
    int localSum = 0;

    if (idx + 3 * blockDim.x < n)
    {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        localSum = a1 + a2 + a3 + a4;
    }

    smem[tid] = localSum;
    __syncthreads();

    // in-place reduction in shared memory using warp unrolling and mask
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride)
            smem[tid] += smem[tid + stride];
        __syncthreads();
    }
    
    // unrolling warp using warp shuffle
    if (tid < 32) {
        volatile int *vsmem = smem;
        for (int offset = 32; offset > 0; offset >>= 1) {
            localSum += __shfl_down_sync(FULL_MASK, localSum, offset);
        }
        vsmem[tid] = localSum;
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}