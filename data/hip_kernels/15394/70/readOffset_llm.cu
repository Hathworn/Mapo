#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffset(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Use shared memory to improve memory access efficiency
    extern __shared__ float sharedA[];
    extern __shared__ float sharedB[];

    if (k < n) {
        sharedA[threadIdx.x] = A[k];
        sharedB[threadIdx.x] = B[k];
        __syncthreads(); // Ensure all threads have loaded data into shared memory
        C[i] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
    }
}