#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_reduction(int *shared_var, int *input_values, int N, int iters)
{
    __shared__ int local_mem[256];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    // Calculate the number of valid threads in this block
    int local_dim = blockDim.x;
    int minThreadInThisBlock = blockIdx.x * blockDim.x;
    int maxThreadInThisBlock = minThreadInThisBlock + (blockDim.x - 1);
    if (maxThreadInThisBlock >= N) {
        local_dim = N - minThreadInThisBlock;
    }

    for (int iter = 0; iter < iters; iter++)
    {
        if (tid < N)
        {
            local_mem[local_tid] = input_values[tid];
        }
        __syncthreads();  // Ensure all threads have written their data

        // Perform the local reduction using parallel reduction technique
        for (int s = local_dim / 2; s > 0; s >>= 1) {
            if (local_tid < s) {
                local_mem[local_tid] += local_mem[local_tid + s];
            }
            __syncthreads();
        }

        if (local_tid == 0)
        {
            atomicAdd(shared_var, local_mem[0]);
        }
        __syncthreads();  // Ensure shared_var update visibility
    }
}