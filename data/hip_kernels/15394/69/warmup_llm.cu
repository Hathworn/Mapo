#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate k only if within bounds
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        unsigned int k = i + offset;
        if (k < n) {
            C[i] = A[k] + B[k]; // Optimize memory access
        }
    }
}