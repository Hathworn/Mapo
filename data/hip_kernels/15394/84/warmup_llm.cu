#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global index directly
    unsigned int global_idx = blockIdx.x * blockDim.x + threadIdx.x + offset;

    // Use branchless operation by adding additional condition in the calculation
    C[global_idx - offset] = (global_idx < n) * (A[global_idx] + B[global_idx]);
}