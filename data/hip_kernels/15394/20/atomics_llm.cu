#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void atomics(int *shared_var, int *values_read, int N, int iters)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N) return;

    // Combine the two atomicAdds to reduce contention and improve performance
    int local_increment = iters + 1;

    // First atomic add captures the initial read value
    values_read[tid] = atomicAdd(shared_var, local_increment) + iters;
}