#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Limit iteration to work on valid data
    if (idx < N)
    {
        // Perform the addition operation once per thread
        C[idx] = A[idx] + B[idx];
    }
}