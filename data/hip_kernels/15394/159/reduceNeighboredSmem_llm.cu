#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighboredSmem(int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];

    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (idx >= n) return;

    // Load data to shared memory
    smem[tid] = g_idata[idx];
    __syncthreads();

    // In-place reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            smem[tid] += smem[tid + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}