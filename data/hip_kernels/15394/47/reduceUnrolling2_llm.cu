#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling2(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // Unrolling 2
    int sum = 0;
    if (idx < n) sum = g_idata[idx]; // Load element
    if (idx + blockDim.x < n) sum += g_idata[idx + blockDim.x]; // Unrolled addition

    idata[tid] = sum; // Save the sum in shared memory

    __syncthreads();

    // In-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        // Synchronize within thread block
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}