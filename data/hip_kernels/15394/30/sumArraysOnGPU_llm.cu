#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // Calculate global thread index for entire grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we don't access out of bounds memory
    if (i < N) C[i] = A[i] + B[i];
}