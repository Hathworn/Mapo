#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU()
{
    // Use thread and block indexing for parallel execution across threads
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) // Only a single thread needs to print
    {
        printf("Hello World from GPU!\n");
    }
}