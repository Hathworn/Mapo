#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iKernel(float *A, float *B, float *C, const int N)
{
    // Use const for repeated calculations to optimize register usage
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Add bounds check to reduce divergence
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}