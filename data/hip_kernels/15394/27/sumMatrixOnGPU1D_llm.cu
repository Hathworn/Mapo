#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Combine 2D threads into a single 1D thread ID
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int totalThreads = gridDim.x * blockDim.x;

    // Use a single loop for better memory access coalescing
    for (int i = idx; i < nx * ny; i += totalThreads)
    {
        MatC[i] = MatA[i] + MatB[i];
    }
}