#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lots_of_float_compute(float *inputs, int N, size_t niters, float *outputs)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t nthreads = gridDim.x * blockDim.x;

    for (; tid < N; tid += nthreads)
    {
        size_t iter;
        float val = inputs[tid];

        // Pre-calculate constants outside the loop
        const float const1 = 1.0f / 3.0f;
        const float const2 = 1.0f / 1.037f;
        const float const3 = 1.0f / 0.22f;
        const float offset1 = 5.0f - 101.0f;
        const float offset2 = 102.0f - 103.0f;
        const float offset3 = 104.0f - 105.0f;
        const float offset4 = 106.0f;

        for (iter = 0; iter < niters; iter++)
        {
            // Consolidate operations to reduce instruction count
            val = ((val + offset1) * const1 + offset2) * const2 + offset3;
            val = val * const3 + offset4;
        }

        outputs[tid] = val;
    }
}