#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeOffsetUnroll2(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;  // Calculate 2 block strides
    unsigned int k = i + offset;

    if (k < n)
    {
        // Load data before use to optimize memory access
        float Ai = A[i];
        float Bi = B[i];
        
        C[k] = Ai + Bi;

        if (i + blockDim.x < n)  // Ensure second write is within bounds
        {
            float Ai1 = A[i + blockDim.x];
            float Bi1 = B[i + blockDim.x];
            C[k + blockDim.x] = Ai1 + Bi1;
        }
    }
}