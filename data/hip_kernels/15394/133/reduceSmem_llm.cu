#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmem (int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // set to smem by each thread
    smem[tid] = idata[tid];
    __syncthreads();

    // In-place reduction in shared memory with loop unrolling
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) smem[tid] += smem[tid + s];
        __syncthreads();
    }

    // Unrolling last warp
    if (tid < 32) {
        volatile int *vsmem = smem; // Use volatile to prevent compiler optimization
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}