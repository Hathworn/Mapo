#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_1d(float *in, float *out)
{
    // Optimized shared memory allocation
    extern __shared__ float smem[];

    // index to global memory
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // index to shared memory for stencil calculation
    int sidx = threadIdx.x + RADIUS;

    // Read data from global memory into shared memory
    smem[sidx] = in[idx];

    // read halo part to shared memory
    if (threadIdx.x < RADIUS)
    {
        smem[sidx - RADIUS] = in[idx - RADIUS];
        smem[sidx + BDIM] = in[idx + BDIM];
    }

    // Synchronize (ensure all the data is available)
    __syncthreads();

    // Apply the stencil with unrolled loop for improved performance
    float tmp = 0.0f;
    #pragma unroll
    for (int i = 1; i <= RADIUS; i++)
    {
        tmp += coef[i] * (smem[sidx + i] - smem[sidx - i]);
    }

    // Store the result
    out[idx] = tmp;
}