#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Use shared memory for potentially faster access
    __shared__ float sMatA[32][32];
    __shared__ float sMatB[32][32];

    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        // Load data into shared memory
        sMatA[threadIdx.y][threadIdx.x] = MatA[idx];
        sMatB[threadIdx.y][threadIdx.x] = MatB[idx];
        __syncthreads();  // Ensure both matrices are fully loaded

        // Perform addition using shared memory
        MatC[idx] = sMatA[threadIdx.y][threadIdx.x] + sMatB[threadIdx.y][threadIdx.x];
    }
}