#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global index with offset in shared memory for better performance
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    if (k < n) {
        // Ensure coalesced memory access by reading and writing sequential elements
        float a_val = A[k]; 
        float b_val = B[k];
        C[i] = a_val + b_val;
    }
}