#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_2dfd(float *g_u1, float *g_u2, const int nx, const int iStart, const int iEnd)
{
    // global to line index
    unsigned int ix  = blockIdx.x * blockDim.x + threadIdx.x;

    // smem idx for current point
    unsigned int stx = threadIdx.x + NPAD;
    unsigned int idx  = ix + iStart * nx;

    // shared memory for x dimension
    __shared__ float line[BDIMX + NPAD2];

    // a coefficient related to physical properties
    const float alpha = 0.12f;

    // register for y value
    float yval[9];

    // Cache initial y values
    for (int i = 0; i < 8; i++) yval[i] = g_u2[idx + (i - 4) * nx];

    // skip for the bottom most y value
    int iskip = NPAD * nx;

    // Loop unrolling and shared memory optimization
    for (int iy = iStart; iy < iEnd; iy++)
    {
        // get yval[8] here for next computation
        yval[8] = g_u2[idx + iskip];

        // Efficient shared memory access
        if(threadIdx.x < NPAD)
        {
            line[threadIdx.x]  = g_u2[idx - NPAD];
            line[stx + BDIMX]  = g_u2[idx + BDIMX];
        }

        line[stx] = yval[4];
        __syncthreads();

        // Boundary check and computation
        if (ix < nx - NPAD)
        {
            // Use registers for temporary variables
            float tmp = coef[0] * line[stx] * 2.0f;

            #pragma unroll
            for(int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (line[stx - d] + line[stx + d]);
            }

            #pragma unroll
            for(int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (yval[4 - d] + yval[4 + d]);
            }

            // Efficiently update g_u1 with computed values
            g_u1[idx] = yval[4] + yval[4] - g_u1[idx] + alpha * tmp;
        }

        // Shift y values for next iteration
        yval[0] = yval[1]; yval[1] = yval[2]; yval[2] = yval[3]; yval[3] = yval[4];
        yval[4] = yval[5]; yval[5] = yval[6]; yval[6] = yval[7]; yval[7] = yval[8];

        // Advance global index
        idx  += nx;
        __syncthreads();
    }
}