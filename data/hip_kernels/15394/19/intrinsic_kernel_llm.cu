#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void intrinsic_kernel(float a, float *out, int iters)
{
    // Calculate thread index
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

    // Only one thread performs the operation
    if(tid == 0)
    {
        // Use register to store intermediate value
        float tmp = __powf(a, 2.0f);

        // Loop unrolling
        for (int i = 1; i < iters; i += 4)
        {
            tmp = __powf(a, 2.0f);
            tmp = __powf(a, 2.0f);
            tmp = __powf(a, 2.0f);
            tmp = __powf(a, 2.0f);
        }

        // Write final value to out
        *out = tmp;
    }
}