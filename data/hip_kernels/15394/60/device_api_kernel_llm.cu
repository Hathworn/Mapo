#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_api_kernel(hiprandState *states, float *out, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;
    
    if (tid >= N) return;  // Exit if tid is out of bounds

    hiprandState localState;
    hiprand_init(9384, tid, 0, &localState);  // Use a local state variable

    for (int i = tid; i < N; i += nthreads)
    {
        float rand = hiprand_uniform(&localState);  // Use local state for generating random numbers
        rand = rand * 2;
        out[i] = rand;
    }
}