#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fmad_kernel(double x, double y, double *out)
{
    // Use thread-independent calculation to avoid threadIdx check
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        *out = __fma_rn(x, x, y);  // Using fused multiply-add operation for precision
    }
}