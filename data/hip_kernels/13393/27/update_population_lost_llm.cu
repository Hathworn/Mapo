#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_population_lost(unsigned int *pop, unsigned int rows, unsigned int cols, unsigned int *fixed) {
    unsigned int tid = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x; // Compute global thread ID
    unsigned int totalThreads = gridDim.x * blockDim.x; // Calculate total number of threads

    // Loop over the array with stride equal to totalThreads to utilize all threads
    for (unsigned int i = tid; i < rows * cols; i += totalThreads) {
        if (pop[i] == 1) { // Check if population element is 1
            atomicAdd(fixed, 1); // Atomic operation to prevent race condition
        }
    }
}