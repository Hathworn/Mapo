#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void recombine(unsigned int *p0, unsigned int *p1, unsigned int *off, unsigned int cols) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int boffset = blockIdx.x * blockDim.x + tid;

    if (boffset < cols) { // Only perform operations if within bounds
        unsigned int p = p0[boffset];
        unsigned int q = p1[boffset];
        unsigned int res = off[boffset];

        // Use bitwise operations to compute the result
        res = ((p & ~res) | (q & res));
        off[boffset] = res; // Write back the result
    }
}