#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_population_metadata(unsigned int *pop, unsigned int rows, unsigned int cols, unsigned int *free, unsigned int *lost, unsigned int *fixed) {
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;
    
    __shared__ unsigned int sPop[MAX_THREADS];
    __shared__ unsigned int sMeta[MAX_THREADS];
    
    unsigned int b_offset = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int offset = threadIdx.y * cols + b_offset;
    unsigned int mat_size = rows * cols;
    
    // Initialize shared memory meta data
    sMeta[tid] = (threadIdx.y & 1) == 0 ? ~0 : 0;
    __syncthreads();

    unsigned int i = 0;
    while (i < rows) {
        sPop[tid] = (offset < mat_size) ? pop[offset] : 0;
        __syncthreads();

        // Perform bitwise operations based on thread responsibility
        if (offset < mat_size) {
            unsigned int d = sPop[tid];
            unsigned int e = sPop[tid + ((threadIdx.y & 1) == 0 ? blockDim.x : 0)];
            sMeta[tid] = (threadIdx.y & 1) == 0 ? (sMeta[tid] & (d & e)) : (sMeta[tid] | (d | e));
        }
        __syncthreads();
        
        offset += blockDim.y * cols;
        i += blockDim.y;
    }

    // Hierarchical warp reduction
    for (i = 4; i <= 32; i <<= 1) {
        unsigned int t = tid + (i / 2) * blockDim.x;
        unsigned int masked = threadIdx.y & (i - 1);
        __syncthreads();

        if (masked == 0) {
            sMeta[tid] &= sMeta[t % MAX_THREADS];
        } else if (masked == 1) {
            sMeta[tid] |= sMeta[t % MAX_THREADS];
        }
        __syncthreads();
    }
    
    // Final write-back using a single warp
    if (threadIdx.y == 0) {
        unsigned int fxd = sMeta[threadIdx.x];
        unsigned int lst = ~sMeta[blockDim.x + threadIdx.x];
        free[b_offset] = (fxd | lst);
        fixed[b_offset] = fxd;
        lost[b_offset] = lst;
    }
    __syncthreads();
}