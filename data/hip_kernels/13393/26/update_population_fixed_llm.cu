#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_population_fixed(unsigned int *pop, unsigned int rows, unsigned int cols, unsigned int *fixed) {
    // Calculate unique thread ID
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread ID is within bounds
    if (idx < rows * cols) {
        // Only update if not fixed
        if (fixed[idx] == 0) {
            pop[idx] = 1; // Example operation to update population
        }
    }
}