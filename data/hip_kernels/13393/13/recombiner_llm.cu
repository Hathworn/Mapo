#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void recombiner(double *rands, unsigned int *parents, unsigned int parent_rows, unsigned int parent_cols, unsigned int *off, unsigned int cols, unsigned int seq_offset) {
    // Load random value only if necessary
    unsigned int id_offset = static_cast<unsigned int>(rands[seq_offset + blockIdx.y]);

    // Calculate only once outside conditional block
    unsigned int col_offset = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate parent's starting offset
    unsigned int p0_offset = id_offset * (parent_rows - 1u) / 2u * parent_cols;

    unsigned int p = 0, q = 0, res = 0;

    // Avoid unnecessary __syncthreads() and access only within bounds
    if (col_offset < parent_cols) {
        unsigned int idx = p0_offset + col_offset;
        p = parents[idx];
        q = parents[idx + parent_cols];
    }

    if (col_offset < cols) {
        // Calculate result based on parents and store directly
        res = off[(seq_offset + blockIdx.y) * cols + col_offset];
        res = ((p & ~res) | (q & res));
        off[(seq_offset + blockIdx.y) * cols + col_offset] = res;
    }
}