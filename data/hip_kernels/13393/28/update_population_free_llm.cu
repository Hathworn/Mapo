#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_population_free(unsigned int *fixed, unsigned int *lost, unsigned int *free, unsigned int cols) {
    // Calculating global thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by checking bounds only once
    if (tid < cols) {
        unsigned int f = fixed[tid];
        unsigned int l = lost[tid];
        
        // Combine operations into fewer memory accesses
        free[tid] = f - l;

        // Further optimize computation or memory access if needed
    }
}
```
