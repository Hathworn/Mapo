#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Recursive Implementation of Interleaved Pair Approach
__global__ void reduceInterleaved(int *g_idata, int *g_odata, unsigned int n) {
    // Improved approach: Eliminate bank conflicts
    extern __shared__ int sdata[];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Copy input to shared memory and boundary check
    if (idx < n) {
        sdata[tid] = g_idata[idx];
    } else {
        sdata[tid] = 0;
    }
    
    __syncthreads();

    // in-place reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}