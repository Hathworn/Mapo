#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel using warp shuffle for reduction
__global__ void reduceUnrollingOptimized(int *g_idata, int *g_odata, unsigned int n, unsigned int q)
{
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * q + tid;

    int sum = 0;
    if (idx + blockDim.x * (q - 1) < n)
    {
        // Perform unrolling and load data
        for (int i = 0; i < q; i++)
        {
            sum += g_idata[idx + blockDim.x * i];
        }
    }
    __syncthreads();

    // Perform warp-level reduction
    for (int stride = warpSize / 2; stride > 0; stride /= 2)
    {
        sum += __shfl_down(sum, stride);
    }

    // Store the result for the first thread of the warp
    if (tid % warpSize == 0) g_odata[blockIdx.x * (blockDim.x / warpSize) + tid / warpSize] = sum;

    __syncthreads();

    // Reduce block-level results to the first warp
    if (tid < warpSize)
    {
        volatile int *odata = g_odata;
        for (int stride = warpSize / 2; stride > 0; stride /= 2)
        {
            if (tid < stride)
            {
                odata[tid] += odata[tid + stride];
            }
        }

        // Write final result for this block to global memory
        if (tid == 0) g_odata[blockIdx.x] = odata[0];
    }
}