#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling (double *g_idata, double *g_odata, unsigned int n, unsigned int q)
{
    // set thread ID and global offset
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * q + threadIdx.x;

    // Perform unrolling for q elements
    double sum = 0.0;
    if (idx + blockDim.x * (q - 1) < n) {
        for (int i = 0; i < q; i++) {
            sum += g_idata[idx + blockDim.x * i];
        }
    }

    // Store the sum back into the input array
    g_idata[idx] = sum;
    __syncthreads();

    // In-place reduction with log step optimization
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            g_idata[idx] += g_idata[idx + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = g_idata[idx];
}