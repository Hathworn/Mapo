#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling (int *g_idata, int *g_odata, unsigned int n, unsigned int q)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * q + threadIdx.x;

    // Unroll loop and accumulate
    for (int i = 0; i < q && idx + blockDim.x * i < n; i++) {
        g_idata[idx] += g_idata[idx + blockDim.x * i];
    } 
    __syncthreads();

    // Reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            g_idata[idx] += g_idata[idx + stride];
        }
        __syncthreads(); // Ensure all threads have completed the stride addition
    }

    // Write result for this block to global mem
    if (tid == 0) {
        g_odata[blockIdx.x] = g_idata[idx];
    }
}