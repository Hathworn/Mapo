#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceInterleaved(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (idx >= n) return;
    
    // In-place reduction in global memory
    int sum = g_idata[idx]; // Load data into register
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        if (tid < stride)
        {
            sum += g_idata[idx + stride]; // Use register variable
        }
        __syncthreads();
        g_idata[idx] = sum; // Store partial result back
    }

    // Write result for this block to global memory
    if (tid == 0) 
    {
        g_odata[blockIdx.x] = g_idata[idx];
    }
}