#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceInterleaved(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n) return;

    // Ensure no out-of-bounds access
    int sum = (idx < n) ? g_idata[idx] : 0;

    // perform reduction using shared memory to minimize global memory transactions
    __shared__ int sdata[1024]; // assuming blockDim.x <= 1024
    sdata[tid] = sum;
    __syncthreads();

    // unrolling the last warp to reduce __syncthreads overhead
    for (int stride = blockDim.x / 2; stride > 32; stride /= 2) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    // unroll the last warp explicitly
    if (tid < 32) {
        volatile int* smem = sdata;
        smem[tid] += smem[tid + 32];
        smem[tid] += smem[tid + 16];
        smem[tid] += smem[tid + 8];
        smem[tid] += smem[tid + 4];
        smem[tid] += smem[tid + 2];
        smem[tid] += smem[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}