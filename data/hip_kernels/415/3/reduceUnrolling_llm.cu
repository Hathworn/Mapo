#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // unroll 2
    int sum = 0;
    if (idx < n) {
        sum = g_idata[idx];
        if (idx + blockDim.x < n) {
            sum += g_idata[idx + blockDim.x];
        }
    }
    __syncthreads();

    // in-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        if (tid < stride)
        {
            sum += __shfl_down_sync(0xFFFFFFFF, sum, stride);
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sum;
}