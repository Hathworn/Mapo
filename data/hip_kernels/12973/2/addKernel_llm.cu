#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if (i < N) // Boundary check to avoid accessing out-of-bound memory
    {
        c[i] = a[i] + b[i];
    }
}