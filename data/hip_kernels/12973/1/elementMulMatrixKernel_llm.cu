#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementMulMatrixKernel(double *dev_w, const double *dev_U, const double *dev_V, unsigned int index_row_i, unsigned int index_column_j, unsigned int dim1_U, unsigned int dim1_V)
{
    // Determine global thread index
    unsigned int idx_k = blockIdx.x * blockDim.x + threadIdx.x;

    // Check index range to abort
    if (idx_k >= dim1_U)
        return;

    // Calculate indices for U and V matrices
    unsigned int idx_u_i0 = index_row_i * dim1_U;
    unsigned int idx_v_0j = index_column_j;

    unsigned int idx_u_ik = idx_u_i0 + idx_k;
    unsigned int idx_v_kj = idx_v_0j + idx_k * dim1_V;

    // Perform the element-wise multiplication
    dev_w[idx_k] = dev_U[idx_u_ik] * dev_V[idx_v_kj];
}