#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_deconv_groups() {
    // Use __syncthreads() for better synchronization across threads in a block
    __syncthreads();
}