#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized vec_tanh kernel
__global__ void vec_tanh(int n, double *result, double *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D block and grid for better performance
    if (id < n) {
        result[id] = tanh(x[id]);
    }
}