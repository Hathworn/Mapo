#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_computePSF_phaseNMany_f(int n, int sizePart, int sizeTot, float *kx, float *ky, float *kz, float *pupil, float *phase, float *position, int *sparseIndexEvenDisk, int *sparseIndexOddDisk, float *fft, int many) {
    // Shared memory and register optimization
    extern __shared__ float shared_mem[];
    float *shared_position = shared_mem;
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = idy * gridDim.x * blockDim.x + tid;
    
    // Load position into shared memory for efficiency
    if (tid < many * 3) {
        shared_position[tid] = position[tid];
    }
    __syncthreads();

    if (id < n) {
        int u = id / sizePart;
        int p = id % sizePart;
        
        // Compute x using shared memory
        float x = kx[p] * shared_position[u] + ky[p] * shared_position[u + many] + kz[p] * shared_position[u + 2 * many] + phase[p];
        
        // Use registers for intermediate computations
        float cos_x = cosf(x);
        float sin_x = sinf(x);
        
        // Write results to global memory
        fft[sparseIndexEvenDisk[p] + u * sizeTot] = pupil[p] * cos_x;
        fft[sparseIndexOddDisk[p] + u * sizeTot] = pupil[p] * sin_x;
    }
}