#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_divScalarMany_f(int n, int sizeSubImage, float *result, double *resultDouble, float *x, float *div)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.x * gridDim.x + threadIdx.y * gridDim.x) * blockDim.y;

    // Optimization: Use faster global memory access pattern
    if (id < n)
    {
        int id2 = __float2int_rd(__fdivide_rd(id, sizeSubImage)); // Use integer division for id2
        if (div[id2] != 0.0f)
        {
            float tmpResult = __fdividef(x[id], div[id2]); // Use faster division for float
            result[id] = tmpResult;
            resultDouble[id] = (double)tmpResult; // Eliminate unnecessary initialization
        }
    }
}