#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// The global kernel function for negating a vector
__global__ void vec_negate(int n, double *result, double *x) {
    // Calculate unique thread index across grid
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if within bounds
    if (id < n) {
        result[id] = -x[id]; // Negate the element
    }
}
```
