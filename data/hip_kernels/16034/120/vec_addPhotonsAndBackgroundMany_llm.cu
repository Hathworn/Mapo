#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void vec_addPhotonsAndBackgroundMany(int n, int sizeSubImage, double *output, double *input, double *photonAndBackground) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within bounds
    if (id < n) {
        int id2 = id / sizeSubImage;

        // Simplified computation using pointer arithmetic
        double photonAndBack = photonAndBackground[id2 * 2];
        double background = photonAndBackground[id2 * 2 + 1];
        
        output[id] = input[id] * photonAndBack + background;
    }
}