#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//=== Vector comparison ======================================================

//WARNING : device_sum size should be gridDim.x
__global__ void vec_erf (int n, double *result, double *x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Flattened index calculation
    if (idx < n) {
        result[idx] = erf(x[idx]); // Compute erf only when idx is valid
    }
}
```
