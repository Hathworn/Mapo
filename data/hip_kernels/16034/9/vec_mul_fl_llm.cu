#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_mul_fl (int n, float *result, float *x, float *y)
{
    // Calculate the unique thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Process only if id is within bounds
    if (id < n)
    {
        result[id] = x[id] * y[id];
    }
}