#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel using 2D grid and shared memory to improve data locality and memory coalescing
__global__ void vec_computeModelMany3_scmos(int n, int sizeImage, double *result, double *x, double *amplitude, double *background, double *scmos) 
{
    // Calculate global thread indices
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Calculate unique thread id for computation
    int id = idy * (gridDim.x * blockDim.x) + idx;
    
    if (id < n) 
    {
        // Calculate unique indices for input arrays based on id
        int id2 = id / sizeImage;
        int id3 = id % sizeImage;

        // Perform computation and store in result array
        result[id] = x[id] * amplitude[id2] + background[id3] + scmos[id3];
    }
}