#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_computePSF_signalNsqrtMany_fcrop(int n, int sizeSubImage, int sizeSubImageFull, float *result, float *fft, float divide, int *sparseIndexEvenShift2D, int *sparseIndexOddShift2D)
{
    // Calculate global indices for 2D grid
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    // Calculate unique global ID for linear access
    int id = idy * gridDim.x * blockDim.x + idx;

    // Precompute offsets
    int id2 = id % sizeSubImage;
    int id3 = id / sizeSubImage;
    int id4 = id3 * sizeSubImageFull;

    if (id < n)
    {
        // Fetch and compute only once, improving efficiency
        float evenIndexVal = fft[sparseIndexEvenShift2D[id2] + id4 * 2];
        float oddIndexVal = fft[sparseIndexOddShift2D[id2] + id4 * 2];

        // Precompute division and reduce operations
        float x = evenIndexVal / divide;
        float y = oddIndexVal / divide;

        // Store result directly
        result[id] = sqrtf(x * x + y * y);
    }
}