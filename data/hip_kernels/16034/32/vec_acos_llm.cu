#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize thread index calculation for better performance
__global__ void vec_acos(int n, double *result, double *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        result[id] = acos(x[id]);
    }
}