#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING: device_sum size should be gridDim.x
__global__ void vec_sqrt (int n, double *result, double *x)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // 1D ID calculation
    if (id < n) {
        result[id] = sqrt(x[id]); // Perform square root operation
    }
}