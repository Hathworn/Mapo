#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function
__global__ void vec_eqScalar(int n, double *result, double *x, double y) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Flatten the index calculation
    if (id < n) {
        result[id] = (x[id] == y) ? 1.0 : 0.0; // Assign comparison result
    }
}