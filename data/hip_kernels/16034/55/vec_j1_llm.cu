#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

// Optimized kernel function
__global__ void vec_j1(int n, double *result, double *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Use 1D indexing for simplicity
    if (id < n) {
        result[id] = j1(x[id]);  // Compute j1 for the valid range
    }
}