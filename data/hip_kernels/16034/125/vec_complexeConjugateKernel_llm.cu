#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_complexeConjugateKernel(int n, int sizeInput, float *output, float *input, float *inputKernel) {
    // Calculate the global thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = 2 * (idy * gridDim.x * blockDim.x + idx);
    int id2 = id % (sizeInput * 2);
    
    // Precompute scaling factor
    float scalingFactor = rsqrtf((float)sizeInput);

    // Check bounds
    if (id < n * 2) {
        // Load and scale input
        float real = input[id2] * scalingFactor;
        float imag = input[id2 + 1] * scalingFactor;

        // Calculate complex conjugate
        float tmp = imag * inputKernel[id + 1] + real * inputKernel[id];
        output[id + 1] = imag * inputKernel[id] - real * inputKernel[id + 1];
        output[id] = tmp;
    }
}