#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_computePSF_phaseNManywithOil_f(int n, int sizePart, int sizeTot, float *kx, float *ky, float *kz, float *kz_is_imag, float *kz_oil, float *kz_oil_is_imag, float *pupil, float *phase, float *position, int *sparseIndexEvenDisk, int *sparseIndexOddDisk, float *fft, int many) {
    // Calculate global index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = idy * gridDim.x * blockDim.x + idx;

    if (id < n) {
        // Compute indices based on partition size
        int u = id / sizePart;
        int p = id % sizePart;

        // Pre-compute common term
        float z = kx[p] * position[u] + ky[p] * position[u + many] + phase[p];

        // Calculate x and y values
        float x = z + kz[p] * position[u + 2 * many] - kz_oil[p] * position[u + 3 * many];
        float y = z + kz[p] * position[u + 2 * many] * kz_is_imag[p] - kz_oil[p] * position[u + 3 * many] * kz_oil_is_imag[p];

        // Compute FFT values
        fft[sparseIndexEvenDisk[p] + u * sizeTot] = pupil[p] * cosf(x); // Use cosf for float efficiency
        fft[sparseIndexOddDisk[p] + u * sizeTot] = pupil[p] * sinf(y);  // Use sinf for float efficiency
    }
}