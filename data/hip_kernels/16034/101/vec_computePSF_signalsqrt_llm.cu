#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_computePSF_signalsqrt(int n, double *result, double *real, double *imag, double divide) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique thread index
    if (id < n) { // Ensure within bounds
        double x = real[id] / divide;
        double y = imag[id] / divide;
        result[id] = sqrt(x * x + y * y); // Optimize computation
    }
}