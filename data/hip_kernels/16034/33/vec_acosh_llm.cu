#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_acosh(int n, double *result, double *x)
{
    // Combine index computation to reduce arithmetic operations
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure valid range check
    if (id < n)
    {
        result[id] = acosh(x[id]);
    }
}