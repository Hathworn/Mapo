#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_rcbrt(int n, double *result, double *x)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate 1D thread ID
    if (id < n)
    {
        result[id] = rcbrt(x[id]);  // Compute reciprocal cube root
    }
}