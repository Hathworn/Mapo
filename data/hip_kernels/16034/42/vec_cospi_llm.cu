#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

//===========================================================================
// Vector math (two arguments)
//===========================================================================

// WARNING : device_sum size should be gridDim.x
__global__ void vec_cospi(int n, double *result, double *x) {
    // Calculate a 1D linear index from 2D grid and block indices for simplicity
    int id = blockIdx.x * blockDim.x + threadIdx.x + 
             blockIdx.y * gridDim.x * blockDim.x + threadIdx.y * blockDim.x;

    // Ensure index is within bounds before calculating
    if (id < n) {
        result[id] = cospi(x[id]); // Compute cosine of pi times element
    }
}