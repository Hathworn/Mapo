#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_computePSF_signalNsqrt(int n, double *result, double *fft, double divide, int *sparseIndexEvenShift2D, int *sparseIndexOddShift2D) {
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.x * gridDim.x) + (threadIdx.y * blockDim.x);

    // Ensure ID is within range
    if (id < n) {
        // Retrieve and normalize the values
        double x = fft[sparseIndexEvenShift2D[id]] / divide;
        double y = fft[sparseIndexOddShift2D[id]] / divide;
        
        // Calculate result (magnitude)
        result[id] = sqrt(x * x + y * y);
    }
}