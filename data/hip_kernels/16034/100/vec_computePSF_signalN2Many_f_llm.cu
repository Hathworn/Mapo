#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel for coalesced memory access and reduce index calculations
__global__ void vec_computePSF_signalN2Many_f(int n, int sizePart, int sizeTot, float divide, int* sparseIndexEvenShiftOutput, int* sparseIndexOddShiftOutput, float* fft, int* sparseIndexEvenShiftOutputNext, int* sparseIndexOddShiftOutputNext, float* psffft)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use stride loop to process all elements within grid
    for (int id = idx; id < n; id += stride) {
        int u = id / sizePart;
        int p = id % sizePart;

        // Cache indices for coalesced access
        int evenIndex = sparseIndexEvenShiftOutput[p] + sizeTot * u;
        int oddIndex = sparseIndexOddShiftOutput[p] + sizeTot * u;
        int psfIndexEven = sparseIndexEvenShiftOutputNext[p] + 2 * sizePart * u;
        int psfIndexOdd = sparseIndexOddShiftOutputNext[p] + 2 * sizePart * u;

        float x = fft[evenIndex] / divide;
        float y = fft[oddIndex] / divide;

        // Store results
        psffft[psfIndexEven] = x * x + y * y;
        psffft[psfIndexOdd] = 0;
    }
}