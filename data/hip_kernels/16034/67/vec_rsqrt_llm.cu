#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize the kernel function for better performance and readability
__global__ void vec_rsqrt(int n, double *result, double *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // 1D indexing for better utilization
    if (id < n) {
        result[id] = rsqrt(x[id]);
    }
}