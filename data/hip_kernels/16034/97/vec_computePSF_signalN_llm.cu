#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_computePSF_signalN(int n, double *result, double divide, int *sparseIndexEvenShiftOutput, int *sparseIndexOddShiftOutput, double *fft) {
    int id = threadIdx.x + blockIdx.x * blockDim.x + 
             (threadIdx.y + blockIdx.y * blockDim.y) * gridDim.x * blockDim.x;

    // Optimize by eliminating redundant computation of `id`
    if (id < n) {
        double x = fft[sparseIndexEvenShiftOutput[id]] / divide;
        double y = fft[sparseIndexOddShiftOutput[id]] / divide;
        result[id] = x * x + y * y;
    }
}