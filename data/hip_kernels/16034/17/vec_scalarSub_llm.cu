#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for subtracting a scalar from a vector
__global__ void vec_scalarSub(int n, double *result, double x, double *y) {
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x; 

    // Ensure within bounds
    if (id < n) {
        result[id] = x - y[id]; // Perform subtraction
    }
}