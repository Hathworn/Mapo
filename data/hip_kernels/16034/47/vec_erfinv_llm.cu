#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_erfinv (int n, double *result, double *y)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Ensure id is within bounds
    if (id < n)
    {
        // Calculate inverse error function
        result[id] = erfinv(y[id]);
    }
}