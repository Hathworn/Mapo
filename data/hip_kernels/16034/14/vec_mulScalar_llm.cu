#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING : device_sum size should be gridDim.x
__global__ void vec_mulScalar(int n, double *result, double *x, double y) {
    // Combining thread and block indices to calculate the global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * gridDim.x * blockDim.x * blockDim.y) + threadIdx.y * blockDim.x;
    
    if (id < n) {
        // Perform vector-scalar multiplication
        result[id] = x[id] * y;
    }
}