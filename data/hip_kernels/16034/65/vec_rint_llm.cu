#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to round input to the nearest integer value in floating-point
__global__ void vec_rint(int n, double *result, double *x) {
    // Calculate 1D global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Only process valid array index
    if (id < n) {
        result[id] = rint(x[id]);
    }
}