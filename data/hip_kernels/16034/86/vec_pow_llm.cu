#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_pow (int n, double *result, double *x, double *y)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified thread indexing
    if (id < n) 
    {
        result[id] = pow(x[id], y[id]);  // Optimized by removing unnecessary calculations
    }
}