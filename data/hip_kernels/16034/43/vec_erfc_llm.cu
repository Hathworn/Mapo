#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

// Optimized kernel using 1D block and grid mapping
__global__ void vec_erfc(int n, double *result, double *x) {
    // Compute global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check boundary condition
    if (id < n) {
        result[id] = erfc(x[id]);
    }
}