#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_computeLocalMaxima(int n, int *localMaxima, int sizeImage, float *input, int sizeFilt, int edgeSizeReject)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = idy * gridDim.x * blockDim.x + idx;

    if (id < n)
    {
        localMaxima[id] = 0;
        int sizeSquare = sizeImage * sizeImage;
        int z = id / sizeSquare;
        int p = id % sizeSquare;
        int x = p / sizeImage;
        int y = p % sizeImage;
        int sizeReject = max(sizeFilt, edgeSizeReject);

        if ((x >= sizeReject) && (y >= sizeReject) && (x + sizeReject < sizeImage) && (y + sizeReject < sizeImage))
        {
            bool isLocalMaxima = true;  // use boolean for better readability
            for (int a = -sizeFilt; a <= sizeFilt; a++)
            {
                for (int aa = -sizeFilt; aa <= sizeFilt; aa++)  // corrected variable name from `a` to `aa`
                {
                    int neighbor = z * sizeSquare + (x + a) * sizeImage + (y + aa);
                    if (input[id] < input[neighbor])
                    {
                        isLocalMaxima = false;
                        break;  // break from the inner loop
                    }
                }
                if (!isLocalMaxima)
                {
                    break;  // break from the outer loop when not a local maxima
                }
            }
            if (isLocalMaxima)
            {
                localMaxima[id] = 1;
            }
        }
    }
}