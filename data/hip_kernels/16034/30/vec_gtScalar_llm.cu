#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel 
__global__ void vec_gtScalar(int n, double *result, double *x, double y) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified ID computation
    if (id < n) {
        result[id] = (x[id] > y) ? 1.0 : 0.0;  // Use ternary operation for comparison
    }
}