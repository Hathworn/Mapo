#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_asin(int n, double *result, double *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified global ID calculation
    if (id < n) {
        result[id] = asin(x[id]);
    }
}