#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel: Use 1D grid and 1D block for simplicity and better performance
__global__ void vec_divScalar(int n, double *result, double *x, double y)
{
    // Calculate global index for 1D grid
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure id is within bounds
    if (id < n)
    {
        result[id] = x[id] / y;
    }
}