#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void vec_sinpi(int n, double *result, double *x)
{
    // Calculate one-dimensional global thread index directly
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread id is within bounds
    if (id < n)
    {
        // Direct computation of sinpi using input array
        result[id] = sinpi(x[id]);
    }
}