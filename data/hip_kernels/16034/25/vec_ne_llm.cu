#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_ne (int n, double *result, double *x, double *y)
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x + threadIdx.y * gridDim.x;

    // Check boundary and calculate result
    if (id < n)
    {
        result[id] = (x[id] != y[id]) ? 1.0 : 0.0;
    }
}