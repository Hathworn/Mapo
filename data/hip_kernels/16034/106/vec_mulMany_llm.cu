#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to perform vector multiplication
__global__ void vec_mulMany(int n, int sizeKernel, double *result, double *x, double *kernel)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bounds and perform multiplication
    if (id < n) {
        int id2 = id % sizeKernel; // Efficient modulo operation
        result[id] = x[id] * kernel[id2];
    }
}