#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimization: merged declaration sections and removed redundant declarations.

// WARNING: device_sum size should be gridDim.x
__global__ void vec_computeModelMany2(int n, int sizeImage, double* result, double* x, double* amplitude, double* background) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Total threads in grid

    // Loop over elements with stride to cover all elements in large data
    for (int id = tid; id < n; id += stride) {
        int id2 = id / sizeImage; // Calculate id2 considering sizeImage
        // Perform computation
        result[id] = x[id] * amplitude[id2] + background[id2];
    }
}