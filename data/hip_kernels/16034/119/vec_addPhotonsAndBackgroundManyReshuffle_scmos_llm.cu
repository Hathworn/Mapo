#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_addPhotonsAndBackgroundManyReshuffle_scmos (int n, int sizeSubImage,int numberPSFperModel,double *output, double *input, double *photonAndBackground, double * scmos)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the thread ID is within bounds
    if (id < n)
    {
        // Use integer division and remainder to calculate indices and positions more efficiently
        int idPSF = id / sizeSubImage;
        int idModel = idPSF / numberPSFperModel;
        int idposit = idPSF % numberPSFperModel;
        int idoffset = id % sizeSubImage;

        // Calculate reshuffled index using simplified calculation
        int idreshuffle = idModel * sizeSubImage +
                          idposit * sizeSubImage * (n / (sizeSubImage * numberPSFperModel)) +
                          idoffset;

        // Perform output calculation
        output[idreshuffle] = input[id] * photonAndBackground[idPSF * 2] +
                              photonAndBackground[idPSF * 2 + 1] + scmos[id];
    }
}