#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_addPhotonsAndBackgroundMany_scmos (int n, int sizeSubImage, double *output, double *input, double *photonAndBackground, double *scmos)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = idy * gridDim.x * blockDim.x + idx;
    
    // Ensure index is within bounds
    if (id < n) 
    {
        int id2 = id / sizeSubImage;
        double photonBg1 = photonAndBackground[id2 * 2];       // Cache value to reduce global memory access
        double photonBg2 = photonAndBackground[id2 * 2 + 1];   // Cache value to reduce global memory access
        output[id] = input[id] * photonBg1 + photonBg2 + scmos[id];
    }
}