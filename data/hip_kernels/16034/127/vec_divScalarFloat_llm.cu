#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

// Optimized kernel for scalar division
__global__ void vec_divScalarFloat(int n, float *result, float *x, float y) {
    // Calculate global thread ID
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform bounds checking
    if (globalId < n) {
        // Divide vector element by scalar
        result[globalId] = x[globalId] / y;
    }
}