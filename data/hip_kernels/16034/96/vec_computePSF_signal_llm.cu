#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING : device_sum size should be gridDim.x
__global__ void vec_computePSF_signal(int n, double *result, double *real, double *imag, double divide) {
    // Use shared memory for faster access
    __shared__ double shared_real[1024];
    __shared__ double shared_imag[1024];

    int tx = threadIdx.x;
    int idx = tx + blockIdx.x * blockDim.x;
    int ty = threadIdx.y;
    int idy = ty + blockIdx.y * blockDim.y;
    int id = idy * gridDim.x * blockDim.x + idx;

    // Load data into shared memory
    if (id < n) {
        shared_real[tx] = real[id];
        shared_imag[tx] = imag[id];
    }

    __syncthreads();

    if (id < n) {
        double x = shared_real[tx] / divide; // Optimized division
        double y = shared_imag[tx] / divide; // Optimized division
        result[id] = x * x + y * y; // Compute signal
    }
}