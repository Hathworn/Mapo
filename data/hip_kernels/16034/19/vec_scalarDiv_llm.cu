#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_scalarDiv (int n, double *result, double  x, double  *y)
{
    // Calculate global thread ID in one step for simplicity
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * blockDim.y + threadIdx.y * gridDim.x * blockDim.x;
    
    // Check if the current thread ID is within the bounds
    if (id < n)
    {
        result[id] = x / y[id]; // Calculate division
    }
}