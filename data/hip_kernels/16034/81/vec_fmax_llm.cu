#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

// Optimized kernel
__global__ void vec_fmax(int n, double *result, double *x, double *y)
{
    // Use 1D flattening for grid and block
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check within bounds
    if (id < n)
    {
        // Perform vector max operation
        result[id] = fmax(x[id], y[id]);
    }
}