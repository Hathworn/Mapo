#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING : device_sum size should be gridDim.x
__global__ void vec_initIndex(int n, int *index)
{
    // Calculate flattened thread index in grid
    int id = blockIdx.x * blockDim.x + threadIdx.x + 
             (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x;

    // Check bounds and initialize index
    if (id < n)
    {
        index[id] = id;
    }
}