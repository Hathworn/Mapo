#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_ceil(int n, double *result, double *x)
{
    // Calculate 1D global thread index within the grid
    int id = blockIdx.x * blockDim.x + threadIdx.x + 
             (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x;

    // Check boundary condition for safe access
    if (id < n)
    {
        // Perform ceiling operation
        result[id] = ceil(x[id]);
    }
}