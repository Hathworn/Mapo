#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_exp2 (int n, double *result, double *x)
{
    // Compute unique thread index for 1D grid
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        // Calculate exp2 for the input at this specific index
        result[id] = exp2(x[id]);
    }
}