#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize thread index calculations for better performance
__global__ void vec_tan (int n, double *result, double *x)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x + 
             (blockIdx.y * blockDim.x * gridDim.x) + threadIdx.y * blockDim.x;
    if (id < n)
    {
        result[id] = tan(x[id]);
    }
}