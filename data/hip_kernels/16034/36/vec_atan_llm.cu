#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_atan (int n, double *result, double *x)
{
    // Calculate unique thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + 
             (blockIdx.y * blockDim.x * gridDim.x) + 
             (threadIdx.y * gridDim.x * blockDim.x);

    // Ensure the thread operates within bounds
    if (id < n)
    {
        result[id] = atan(x[id]);
    }
}