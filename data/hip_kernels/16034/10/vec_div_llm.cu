#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_div (int n, double *result, double *x, double *y)
{
    // Use a single index calculation to reduce redundant operations
    int id = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.x * gridDim.x) + (threadIdx.y * blockDim.x);
    
    // Use if condition to avoid out-of-bounds access
    if (id < n)
    {
        result[id] = x[id] / y[id];
    }
}