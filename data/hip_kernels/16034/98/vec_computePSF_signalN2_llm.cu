#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function: Compute PSF signal
__global__ void vec_computePSF_signalN2 (int n, double divide, int *sparseIndexEvenShiftOutput, int *sparseIndexOddShiftOutput, double *fft, int *sparseIndexEvenShiftOutputNext, int *sparseIndexOddShiftOutputNext, double *psffft)
{
    // Calculate unique thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (id < n)
    {
        // Load inputs from global memory
        int indexEven = sparseIndexEvenShiftOutput[id];
        int indexOdd = sparseIndexOddShiftOutput[id];

        // Division operation
        double x = fft[indexEven] / divide;
        double y = fft[indexOdd] / divide;

        // Compute output and store in global memory
        psffft[sparseIndexEvenShiftOutputNext[id]] = x * x + y * y;
        psffft[sparseIndexOddShiftOutputNext[id]] = 0.0;
    }
}