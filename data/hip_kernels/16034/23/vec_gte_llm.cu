#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using blockDim.x index calculation for 1D block
__global__ void vec_gte (int n, double *result, double  *x, double  *y)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // 1D block thread index
    if (id < n)
    {
        result[id] = (x[id] >= y[id]) ? 1.0 : 0.0;
    }
}