#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_exp (int n, double *result, double *x)
{
    // Optimize by calculating a flat index directly
    int id = blockIdx.x * blockDim.x + threadIdx.x + 
             (blockIdx.y * blockDim.y + threadIdx.y) * (blockDim.x * gridDim.x);
    
    // Check bounds and perform computation
    if (id < n) 
    {
        result[id] = exp(x[id]);
    }
}