#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_computePoissonLikelihood(int n, double *result, double *image, double *model)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.x * gridDim.x + threadIdx.y * gridDim.x * blockDim.x);
    
    // Check if thread id is within bounds
    if (id < n)
    {
        double model_val = model[id];
        
        // If model value is greater than 0, calculate result
        if (model_val > 0.0)
        {
            result[id] = model_val - image[id] * log(model_val);
        }
        else
        {
            // Assign a large number if model value is less than or equal to 0
            result[id] = 10000000.0;
        }
    }
}