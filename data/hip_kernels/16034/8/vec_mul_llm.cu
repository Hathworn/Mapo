#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimization of the vec_mul kernel function
__global__ void vec_mul (int n, double *result, double *x, double *y)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate linear index using grid stride
    if (id < n)
    {
        result[id] = x[id] * y[id]; // Multiply vectors if index within bounds
    }
}