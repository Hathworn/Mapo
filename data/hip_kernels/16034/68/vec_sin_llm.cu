#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void vec_sin(int n, double *result, double *x)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate 1D global thread index
    if (id < n) 
    {
        result[id] = sin(x[id]); // Compute sine if within bounds
    }
}