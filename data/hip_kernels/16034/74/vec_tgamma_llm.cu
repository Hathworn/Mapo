#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_tgamma(int n, double *result, double *x)
{
    // Calculate flattened thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (id < n)
    {
        // Compute the gamma function
        result[id] = tgamma(x[id]);
    }
}