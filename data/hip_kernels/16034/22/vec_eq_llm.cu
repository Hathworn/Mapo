#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_eq (int n, double *result, double *x, double *y)
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.x * gridDim.x);
    
    // Check bounds and perform comparison
    if (id < n) {
        result[id] = (x[id] == y[id]) ? 1.0 : 0.0;
    }
}