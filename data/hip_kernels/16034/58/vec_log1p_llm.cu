#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_log1p(int n, double *result, double *x) {
    // Efficient thread id calculation using flattened 1D grid
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if id is within bounds
    if (id < n) {
        result[id] = log1p(x[id]); // Compute log1p for valid id
    }
}