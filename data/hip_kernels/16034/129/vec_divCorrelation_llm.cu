#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_divCorrelation(int n, float *x, int sizeImage, float *varImage, float *varPSF)
{
    // Calculate thread and block index for flattened 2D grid
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = idy * gridDim.x * blockDim.x + idx;
    
    // Continue if within bounds
    if (id < n)
    {
        int lengthImage = sizeImage * sizeImage;
        int positInImage = id % lengthImage;
        int zposit = id / lengthImage;
        
        // Compute division only if non-negative
        float div = varImage[positInImage] * varPSF[zposit];
        if (div > 0) {
            x[id] = x[id] / sqrtf(div); // Use sqrtf for float type
        } else {
            x[id] = -1;
        }
    }
}