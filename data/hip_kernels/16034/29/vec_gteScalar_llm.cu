#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize the thread index calculation and improve memory access pattern.
__global__ void vec_gteScalar (int n, double *result, double *x, double y)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = (x[id] >= y) ? 1.0 : 0.0;
    }
}