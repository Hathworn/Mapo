#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING : device_sum size should be gridDim.x
__global__ void vec_normcdfinv (int n, double *result, double *y)
{
    // Optimize calculation of id using 1D linear indexing
    int id = blockIdx.x * blockDim.x + threadIdx.x; 
    
    // Ensure access within bounds
    if (id < n) 
    {
        // Calculate inverse of standard normal cumulative distribution function
        result[id] = normcdfinv(y[id]);
    }
}