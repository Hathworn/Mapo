#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_scalarMul (int n, double *result, double x, double *y)
{
    // Calculate unique thread ID for 1D grid
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure valid index within bounds
    if (id < n)
    {
        result[id] = x * y[id];  // Perform scalar multiplication
    }
}