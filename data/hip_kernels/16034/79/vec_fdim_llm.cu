#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_fdim(int n, double *result, double *x, double *y) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplify thread index calculation
    if (id < n) {
        result[id] = fdim(x[id], y[id]); // Compute positive difference
    }
}