#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_eraseNonLocalMaxima(int n, float *input, int *localMaxima)
{
    // Use shared memory to reduce global memory accesses
    __shared__ int sharedLocalMaxima[1024];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = idy * gridDim.x * blockDim.x + idx;

    // Ensure threads do not process out-of-range elements
    if (id < n) {
        sharedLocalMaxima[threadIdx.x] = localMaxima[id];
        __syncthreads();

        // Use shared memory in condition check
        if (sharedLocalMaxima[threadIdx.x] == 0) {
            input[id] = -1;
        }
    }
}