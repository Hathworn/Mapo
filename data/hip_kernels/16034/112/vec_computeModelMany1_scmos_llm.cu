#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize the kernel function
__global__ void vec_computeModelMany1_scmos(int n, int sizeImage, double *result, double *x, double *amplitude, double background, double *scmos)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Flatten block and thread indices
    int id2 = blockIdx.y;  // Use blockIdx.y instead of recalculating id2
    int id3 = id % sizeImage;  // Get id3 from flattened id
    if (id < n)
    {
        result[id] = x[id] * amplitude[id2] + background + scmos[id3];
    }
}
```
