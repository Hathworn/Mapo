#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_hypot(int n, double *result, double *x, double *y) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Optimize indexing for 1D grid
    if (id < n) {
        result[id] = hypot(x[id], y[id]);
    }
}