#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

// Vector arithmetic, scalar arithmetic, and other mathematical operations 
// truncated for brevity, assuming they are present before or after.

__global__ void vec_double2float(int n, float *output, double *input) {
    // Calculate global thread ID for 1D grid with 2D block.
    int id = blockIdx.x * blockDim.x + threadIdx.x + (blockDim.x * gridDim.x) * (blockIdx.y * blockDim.y + threadIdx.y);
    
    // Perform conversion for valid id.
    if (id < n) {
        output[id] = static_cast<float>(input[id]);
    }
}