#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_computePSF_phaseN (int n, double *kx, double *ky, double *kz, double *pupil, double *phase, double dx, double dy, double dz, int *sparseIndexEvenDisk, int *sparseIndexOddDisk, double *fft)
{
    // Calculate unique thread ID more efficiently
    int id = threadIdx.x + (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x;
    
    // Only process within bounds
    if (id < n) {
        double x = kx[id] * dx + ky[id] * dy + kz[id] * dz + phase[id];
        
        // Compute FFT contributions concurrently
        double cos_val = cos(x);
        double sin_val = sin(x);
        
        fft[sparseIndexEvenDisk[id]] = pupil[id] * cos_val;
        fft[sparseIndexOddDisk[id]] = pupil[id] * sin_val;
    }
}