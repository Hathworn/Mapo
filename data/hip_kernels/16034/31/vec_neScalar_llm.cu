#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING : device_sum size should be gridDim.x
__global__ void vec_neScalar(int n, double *result, double *x, double y)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure thread ID is within bounds
    if (id < n)
    {
        result[id] = (x[id] != y) ? 1.0 : 0.0;
    }
}
```
