#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize the kernel by using 1D grid and block for more efficient execution
__global__ void vec_asinh(int n, double *result, double *x)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Flatten the block and grid dimensions into 1D index
    if (id < n)
    {
        result[id] = asinh(x[id]);  // Compute asinh for the valid index
    }
}