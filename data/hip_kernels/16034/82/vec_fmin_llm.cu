#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

// Optimize global kernel function to use 1D grid only
__global__ void vec_fmin(int n, double *result, double *x, double *y) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D indexing
    if (id < n) {
        result[id] = fmin(x[id], y[id]); // Perform vector minimum calculation
    }
}