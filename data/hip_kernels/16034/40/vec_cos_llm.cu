#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_cos (int n, double *result, double  *x)
{
    // Use 1D indexing for simplicity and fewer calculations
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        result[id] = cos(x[id]);
    }
}