#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_exp10 (int n, double *result, double  *x)
{
    // Optimize by using a single index calculation
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a single-dimensional grid for better performance
    if (id < n)
    {
        result[id] = exp10(x[id]);
    }
}