#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function for vector addition with scalar
__global__ void vec_addScalar (int n, double *result, double *x, double y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for loop

    for (int id = idx; id < n; id += stride) // Loop over elements within grid
    {
        result[id] = x[id] + y; // Perform addition with scalar
    }
}