#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_testkernel (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = idy * gridDim.x * blockDim.x + idx;

    // Check if id is within bounds for safety
    if (id < n)
    {
        double temp_x = x[id]; // Load value from global memory
        double temp_y = y[id]; // Load value from global memory
        double precomputedValue = sqrt(pow(3.14159, id)); // Precompute sqrt value
        for (int j = 0; j < 100; j++)
        {
            result[id] = temp_x * temp_y + precomputedValue; // Use temporary variables
        }
    }
}