#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_computePSF_signalNsqrtMany (int n, int sizeSubImage, double *result, double *fft, double divide, int *sparseIndexEvenShift2D, int *sparseIndexOddShift2D)
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x;
    
    // Check if the thread is within bounds
    if (id < n)
    {
        int id2 = id % sizeSubImage;
        int id3 = id / sizeSubImage;

        // Fetch values using precomputed indexes
        double x = fft[sparseIndexEvenShift2D[id2] + id3 * sizeSubImage * 2] / divide;
        double y = fft[sparseIndexOddShift2D[id2] + id3 * sizeSubImage * 2] / divide;

        // Store the result of the computation
        result[id] = sqrt(x * x + y * y);
    }
}