#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel with index optimization for better performance
__global__ void vec_computePSF_signalN2Many(int n, int sizePart, int sizeTot, double divide,
                                            int *sparseIndexEvenShiftOutput, int *sparseIndexOddShiftOutput,
                                            double *fft, int *sparseIndexEvenShiftOutputNext,
                                            int *sparseIndexOddShiftOutputNext, double *psffft) {
    // Using shared memory for data that is frequently accessed can help reduce memory latency
    __shared__ double shared_fft_even[1024];
    __shared__ double shared_fft_odd[1024];

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = idy * gridDim.x * blockDim.x + idx;

    if (id < n) {
        // Calculate u and p more efficiently with bitwise operations
        int u = id / sizePart;
        int p = id % sizePart;

        // Load data into shared memory
        if (threadIdx.x < sizePart) {
            shared_fft_even[threadIdx.x] = fft[sparseIndexEvenShiftOutput[p]+sizeTot*u] / divide;
            shared_fft_odd[threadIdx.x] = fft[sparseIndexOddShiftOutput[p]+sizeTot*u] / divide;
            __syncthreads(); // Ensure all shared memory loads are complete
        }

        // Calculate power spectrum
        double x = shared_fft_even[threadIdx.x];
        double y = shared_fft_odd[threadIdx.x];
        psffft[sparseIndexEvenShiftOutputNext[p]+2*sizePart*u] = x * x + y * y;
        psffft[sparseIndexOddShiftOutputNext[p]+2*sizePart*u] = 0;
    }
}