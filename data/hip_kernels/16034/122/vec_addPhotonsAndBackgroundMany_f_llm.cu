#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimization
__global__ void vec_addPhotonsAndBackgroundMany_f(int n, int sizeSubImage, float *output, float *input, float *photonAndBackground) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = idy * gridDim.x * blockDim.x + idx;

    if (id < n) {
        // Calculate id2 to avoid repeated division operations
        int id2 = __float2int_rd(__fdiv_rd(static_cast<float>(id), static_cast<float>(sizeSubImage)));
        // Update output using precomputed id2
        output[id] = input[id] * photonAndBackground[id2 * 2] + photonAndBackground[id2 * 2 + 1];
    }
}