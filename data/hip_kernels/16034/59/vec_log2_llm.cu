#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize by using 1D grid and 1D block for better memory coalescing
__global__ void vec_log2(int n, double *result, double *x)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Avoid unnecessary computations with direct access
    if (id < n) 
    {
        result[id] = log2(x[id]);
    }
}