#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize the kernel by using 1D block and grid
__global__ void vec_lteScalar (int n, double *result, double *x, double y)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique thread ID
    if (id < n) // Check boundary condition
    {
        result[id] = (x[id] <= y) ? 1.0 : 0.0; // Conditional assignment
    }
}