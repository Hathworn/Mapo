#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING : device_sum size should be gridDim.x
__global__ void vec_remainder(int n, double *result, double *x, double *y) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Combine thread and block indices for 1D
    if (id < n) {
        result[id] = remainder(x[id], y[id]); // Compute the remainder
    }
}