#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access pattern and reduce computations
__global__ void vec_computePSF_phaseNwithOil (int n, double  *kx, double  *ky, double  *kz,double  *kz_is_imag, double  *kz_oil,double  *kz_oil_is_imag, double  *pupil, double  *phase,double dx, double dy, double dz, double dz_oil, int *sparseIndexEvenDisk, int *sparseIndexOddDisk, double *fft) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = idy * gridDim.x * blockDim.x + idx;
    
    if (id < n) {
        // Precompute recurring values
        double kx_dx = kx[id] * dx;
        double ky_dy = ky[id] * dy;
        double z_phase = kx_dx + ky_dy + phase[id];
        double dz_component = kz[id] * dz;
        double dz_oil_component = kz_oil[id] * dz_oil;
        double pupil_val = pupil[id];
        
        // Compute x and y using precomputed values
        double x = z_phase + dz_component - dz_oil_component;
        double y = z_phase + dz_component * kz_is_imag[id] - dz_oil_component * kz_oil_is_imag[id];

        // Write results to the FFT array
        fft[sparseIndexEvenDisk[id]] = pupil_val * cos(x);
        fft[sparseIndexOddDisk[id]] = pupil_val * sin(y);
    }
}