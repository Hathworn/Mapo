#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING : device_sum size should be gridDim.x
__global__ void vec_gt(int n, double *result, double *x, double *y)
{
    // Compute global thread ID using 1D grid and 1D block for simplicity
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (id < n)
    {
        // Compare elements and assign result
        result[id] = (x[id] > y[id]) ? 1.0 : 0.0;
    }
}