#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

// Optimize global kernel function for better performance
__global__ void vec_lgamma(int n, double *result, double *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Flatten 1D grid and block indexing
    if (id < n) {
        result[id] = lgamma(x[id]);
    }
}