#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize by caching grid/block dimensions and thread index calculations
__global__ void vec_nextafter (int n, double *result, double  *x, double  *y)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    // Reduced computation for global index
    int block_size_x = blockDim.x * gridDim.x;
    int id = idy * block_size_x + idx;

    if (id < n)
    {
        result[id] = nextafter(x[id], y[id]);
    }
}