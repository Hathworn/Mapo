#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_cbrt(int n, double *result, double *x) 
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds to prevent accessing invalid memory
    if (id < n) 
    {
        // Perform cube root operation
        result[id] = cbrt(x[id]);
    }
}