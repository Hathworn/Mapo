#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel to perform bounds check and calculation efficiently
__global__ void vec_computeModelMany1 (int n, int sizeImage, double *result, double  *x, double  *amplitude,double background)
{
    // Calculate flattened unique thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + 
             (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x;

    // Check if id is within bounds
    if (id < n)
    {
        // Compute model using shared index
        result[id] = x[id] * amplitude[id / sizeImage] + background;
    }
}