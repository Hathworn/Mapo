#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Macro definition here

extern "C"

// Optimized global kernel function
__global__ void vec_floor(int n, double *result, double *x) {
    // Calculate unique thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + 
             blockIdx.y * blockDim.x * gridDim.x + threadIdx.y * blockDim.x;
    
    // Use a more efficient read pattern
    if (id < n) {
        result[id] = floor(x[id]);  // Perform floor operation
    }
}