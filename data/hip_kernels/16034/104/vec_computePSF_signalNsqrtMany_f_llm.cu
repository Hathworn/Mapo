#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_computePSF_signalNsqrtMany_f (int n, int sizeSubImage, float *result, float *fft, float divide, int *sparseIndexEvenShift2D, int *sparseIndexOddShift2D) {
    // Use shared memory for better data reuse
    __shared__ float sharedFFT[1024];  // Adjust size as needed

    float x, y;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = idy * gridDim.x * blockDim.x + idx;
    int id2 = id % sizeSubImage;
    int id3 = id / sizeSubImage;

    if (id < n) {
        // Load data into shared memory
        int evenIndex = sparseIndexEvenShift2D[id2] + id3 * sizeSubImage * 2;
        int oddIndex = sparseIndexOddShift2D[id2] + id3 * sizeSubImage * 2;
        
        if(threadIdx.x < sizeSubImage * 2) {
            sharedFFT[threadIdx.x] = fft[evenIndex + threadIdx.x] / divide;
            if(threadIdx.x + 1 < sizeSubImage * 2) {
                sharedFFT[threadIdx.x + 1] = fft[oddIndex + threadIdx.x] / divide;
            }
        }

        __syncthreads();  // Ensure all loads are complete

        // Read from shared memory
        x = sharedFFT[sparseIndexEvenShift2D[id2]];
        y = sharedFFT[sparseIndexOddShift2D[id2]];
        result[id] = sqrtf(x * x + y * y);
    }
}