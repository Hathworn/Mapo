#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_computeModelMany3(int n, int sizeImage, double *result, double *x, double *amplitude, double *background)
{
    // Optimize indexing calculation for better performance
    int id = blockIdx.y * gridDim.x * blockDim.x * blockDim.y + blockIdx.x * blockDim.x * blockDim.y 
             + threadIdx.y * blockDim.x + threadIdx.x;
    if (id < n)
    {
        int id2 = id / sizeImage;
        int id3 = id % sizeImage;
        
        // Perform computation
        result[id] = x[id] * amplitude[id2] + background[id3];
    }
}