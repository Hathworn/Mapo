#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING : device_sum size should be gridDim.x
__global__ void vec_erfcinv(int n, double *result, double *y) {
    // Calculate unique thread index
    int id = threadIdx.x + blockIdx.x * blockDim.x + (threadIdx.y + blockIdx.y * blockDim.y) * gridDim.x * blockDim.x;

    // Ensure thread operates within bounds
    if (id < n) {
        // Perform inverse complementary error function calculation
        result[id] = erfcinv(y[id]);
    }
}