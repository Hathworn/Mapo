#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING : device_sum size should be gridDim.x
__global__ void vec_trunc(int n, double *result, double *x)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int id = blockIdx.y * gridDim.x * blockDim.x + idx;

    // Check if thread id is within bounds.
    if (id < n)
    {
        result[id] = trunc(x[id]);
    }
}