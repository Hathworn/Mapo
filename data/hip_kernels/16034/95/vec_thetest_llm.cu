#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING : device_sum size should be gridDim.x
__global__ void vec_thetest(int n, float *kz_is_imag)
{
    // Calculate the global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x + threadIdx.y * gridDim.x * blockDim.x;

    // Ensure thread ID is within bounds
    if (id < n)
    {
        // Output the value at the given index
        printf("kz -> %d  %f\n", id, kz_is_imag[id]);
    }
}