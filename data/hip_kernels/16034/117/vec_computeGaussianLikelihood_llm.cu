#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_computeGaussianLikelihood (int n, double *result, double *image, double *model) {
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x;
    if (id < n) {
        // Optimization: Branch divergence reduction
        double mod = model[id];
        result[id] = (mod > 0) ? (mod - image[id]) * (mod - image[id]) : 10000000;
    }
}