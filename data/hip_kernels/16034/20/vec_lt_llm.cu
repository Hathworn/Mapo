#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void vec_lt (int n, double *result, double *x, double *y)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // 1D index for improved mapping
    if (id < n)
    {
        result[id] = (x[id] < y[id]) ? 1.0 : 0.0; // Keep logic for vector comparison
    }
}