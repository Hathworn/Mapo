#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_log10 (int n, double *result, double *x) {
    int id = threadIdx.x + blockIdx.x * blockDim.x; // Flattened thread index for 1D grid
    if (id < n) {
        result[id] = log10(x[id]); // Compute log10 if within bounds
    }
}