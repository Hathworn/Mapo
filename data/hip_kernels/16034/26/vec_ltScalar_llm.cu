#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access by reducing redundant index calculations
__global__ void vec_ltScalar(int n, double *result, double *x, double y) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use 1D indexing to simplify computation
    while (id < n) {
        result[id] = (x[id] < y) ? 1.0 : 0.0;
        id += blockDim.x * gridDim.x;  // Move to next element in grid
    }
}