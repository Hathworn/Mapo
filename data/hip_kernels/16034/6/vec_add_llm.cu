#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_add (int n, double *result, double  *x, double  *y)
{
    // Calculate unique thread index in a simplified manner
    int id = blockIdx.x * blockDim.x + threadIdx.x; 
    if (id < n) // Check if thread is within bounds
    {
        result[id] = x[id] + y[id]; // Perform vector addition
    }
}