#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_j0(int n, double *result, double *x)
{
    // Calculate the thread's unique index within the grid
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if thread index is within bounds
    if (id < n)
    {
        // Perform the computation using the j0 function
        result[id] = j0(x[id]);
    }
}