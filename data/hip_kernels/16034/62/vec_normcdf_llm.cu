#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_normcdf(int n, double *result, double *y) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate 1D global thread index
    if (id < n) {
        result[id] = normcdf(y[id]); // Efficiently compute normcdf in parallel
    }
}