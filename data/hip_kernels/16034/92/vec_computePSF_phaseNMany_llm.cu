#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_computePSF_phaseNMany(int n, int sizePart, int sizeTot, double *kx, double *ky, double *kz, double *pupil, double *phase, double* position, int *sparseIndexEvenDisk, int *sparseIndexOddDisk, double *fft, int many) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = idy * gridDim.x * blockDim.x + idx;

    // Process only within the relevant range
    if (id < n) {
        int u = id / sizePart;
        int p = id % sizePart;
        
        // Compute position indices upfront for brevity
        double posU = position[u];
        double posUM = position[u + many];
        double posU2M = position[u + 2 * many];
        
        // Compute phase using expanded expression
        double x = kx[p] * posU + ky[p] * posUM + kz[p] * posU2M + phase[p];

        // Cache common indices calculations
        int evenIdx = sparseIndexEvenDisk[p] + u * sizeTot;
        int oddIdx = sparseIndexOddDisk[p] + u * sizeTot;

        // Reduce redundant memory accesses by reducing pointer dereferencing
        double pupilP = pupil[p];
        fft[evenIdx] = pupilP * cos(x);
        fft[oddIdx] = pupilP * sin(x);
    }
}