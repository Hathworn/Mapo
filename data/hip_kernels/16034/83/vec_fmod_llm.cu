#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_fmod(int n, double *result, double *x, double *y) {
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + 
             (blockIdx.y * blockDim.x * gridDim.x) + 
             (threadIdx.y * gridDim.x * blockDim.x);
    
    // Compute fmod only for valid indices
    if (id < n) {
        result[id] = fmod(x[id], y[id]);
    }
}