#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_addPhotonsAndBackgroundManyReshuffle (int n, int sizeSubImage,int numberPSFperModel,double *output, double *input, double *photonAndBackground)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Unroll loop to improve memory coalescing
    for(int i = id; i < n; i += stride) {
        int idPSF = i / sizeSubImage;
        int idModel = idPSF / numberPSFperModel;
        int idoffset = i % sizeSubImage;
        int idposit = idPSF % numberPSFperModel;
        int idreshuffle = idModel * sizeSubImage + idposit * sizeSubImage * (n / (sizeSubImage * numberPSFperModel)) + idoffset;
        
        // Use temporary variables for repeated calculations
        double photonBg1 = photonAndBackground[idPSF * 2];
        double photonBg2 = photonAndBackground[idPSF * 2 + 1];
        
        output[idreshuffle] = input[i] * photonBg1 + photonBg2;
    }
}
```
