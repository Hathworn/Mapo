#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_cosh(int n, double *result, double *x)
{
    // Calculate global index for the thread
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds
    if (id < n)
    {
        result[id] = cosh(x[id]); // Use internal cosh function on input array
    }
}