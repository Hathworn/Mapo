#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_sub(int n, double *result, double *x, double *y)
{
    // Calculate the global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + (blockDim.x * gridDim.x) * (blockIdx.y * blockDim.y + threadIdx.y);

    // Ensure the thread works on valid data
    if (id < n)
    {
        // Perform element-wise subtraction
        result[id] = x[id] - y[id];
    }
}