#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize the use of a single blockIdx for better caching
__global__ void vec_erfcx (int n, double *result, double *x)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (id < n)
    {
        result[id] = erfcx(x[id]);
    }
}