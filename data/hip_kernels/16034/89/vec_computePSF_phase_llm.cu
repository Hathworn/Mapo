#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_computePSF_phase(int n, double *realOutput, double *imagOutput, double *kx, double *ky, double *kz, double *pupil, double *phase, double dx, double dy, double dz)
{
    // Calculate global thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int id = idy * gridDim.x * blockDim.x + idx;

    // Ensure thread operates within bounds
    if (id < n)
    {
        // Precompute terms to reduce redundant calculations
        double kx_dx = kx[id] * dx;
        double ky_dy = ky[id] * dy;
        double kz_dz = kz[id] * dz;
        
        // Calculate phase and corresponding outputs
        double x = kx_dx + ky_dy + kz_dz + phase[id];
        double cos_x = cos(x);
        double sin_x = sin(x);

        // Store outputs
        realOutput[id] = pupil[id] * cos_x;
        imagOutput[id] = pupil[id] * sin_x;
    }
}