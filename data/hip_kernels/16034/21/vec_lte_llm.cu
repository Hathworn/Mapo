#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING: device_sum size should be gridDim.x
__global__ void vec_lte(int n, double *result, double *x, double *y)
{
    // Calculate unique thread index in the grid.
    int id = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.x * gridDim.x + threadIdx.y * gridDim.x * blockDim.x);

    // Check bounds before performing operations.
    if (id < n) {
        // Store result of comparison directly.
        result[id] = (x[id] <= y[id]) ? 1.0 : 0.0;
    }
}