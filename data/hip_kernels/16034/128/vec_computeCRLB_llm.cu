#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Global function optimization
__global__ void vec_computeCRLB(int n, int sizeMatrix, double *output, double *input, double h) {
    int sizeSubImage = n / (sizeMatrix * sizeMatrix);
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation for 1D grid

    if (id < n) {
        int p = id / sizeSubImage;
        int positpix = id % sizeSubImage;
        int line = p / sizeMatrix;
        int column = p % sizeMatrix;

        // Use shared memory to reduce global memory access latency
        extern __shared__ double shared_input[];
        shared_input[threadIdx.x] = input[positpix];
        __syncthreads();

        if (shared_input[threadIdx.x] > 0) {
            double d1 = (input[positpix + sizeSubImage * (line * 2 + 2)] - input[positpix + sizeSubImage * (line * 2 + 1)]) / (2 * h);
            double d2 = (input[positpix + sizeSubImage * (column * 2 + 2)] - input[positpix + sizeSubImage * (column * 2 + 1)]) / (2 * h);
            output[id] = (d1 * d2) / shared_input[threadIdx.x];
        } else {
            output[id] = 1e8; // Use scientific notation for readability
        }
    }
}