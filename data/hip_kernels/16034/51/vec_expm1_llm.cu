#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize vec_expm1 kernel function
__global__ void vec_expm1 (int n, double *result, double *x)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (id < n)
    {
        result[id] = expm1(x[id]); // Compute expm1 for each element if within bounds
    }
}