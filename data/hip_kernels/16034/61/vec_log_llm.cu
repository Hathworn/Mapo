#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_log(int n, double *result, double *x) 
{
    // Calculate global thread ID with 1D grid and 1D block
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure id is within the bounds
    if (id < n) 
    {
        result[id] = log(x[id]);
    }
}