#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING : device_sum size should be gridDim.x
__global__ void vec_y0 (int n, double *result, double *x)
{
    // Use a single thread index for better memory coalescing
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensuring the index is within the range
    if (id < n) {
        result[id] = y0(x[id]);
    }
}