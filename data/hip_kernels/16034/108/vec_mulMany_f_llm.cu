#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized Kernel
__global__ void vec_mulMany_f (int n, int sizeKernel, float *result, float *x, float *kernel)
{
    // Calculate global thread ID
    int id = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * threadIdx.y + gridDim.x * blockIdx.y);
    
    // Bulk write with boundary check
    if (id < n) 
    {
        result[id] = x[id] * kernel[id % sizeKernel];
    }
}