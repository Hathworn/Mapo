#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_y1 (int n, double *result, double *x)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x + 
             blockIdx.y * blockDim.x * gridDim.x;

    // Ensure the thread ID is within bounds
    if (id < n)
    {
        // Compute the Bessel function of the second kind for input value
        result[id] = y1(x[id]);
    }
}