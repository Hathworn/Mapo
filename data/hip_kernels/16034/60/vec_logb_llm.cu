#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel with 1D grid and block indexing
__global__ void vec_logb (int n, double *result, double *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify index calculation
    if (id < n) {
        result[id] = logb(x[id]);
    }
}