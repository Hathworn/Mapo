#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// WARNING : device_sum size should be gridDim.x
__global__ void vec_copysign (int n, double *result, double *x, double *y)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate 1D index
    if (id < n) {
        result[id] = copysign(x[id], y[id]); // Perform copysign operation
    }
}