#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"


//WARNING : device_sum size should be gridDim.x
__global__ void vec_computeModelMany2_scmos (int n, int sizeImage, double *result, double  *x, double  *amplitude, double *background, double  *scmos)
{
    // Calculate global thread index
    int id = blockIdx.y * gridDim.x * blockDim.x * blockDim.y + blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    
    // Ensure thread is within bounds
    if (id < n)
    {
        // Optimize index calculations
        int id2 = id / sizeImage;
        int id3 = id % sizeImage;
        
        // Compute the model
        result[id] = x[id] * amplitude[id2] + background[id2] + scmos[id3];
    }
}