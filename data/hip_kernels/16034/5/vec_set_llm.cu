#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function with block-level parallelism
__global__ void vec_set(int n, double *result, double value)
{
    // Calculate global index using blockIdx and threadIdx
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure within bounds before storing
    if (id < n) {
        result[id] = value;
    }
}