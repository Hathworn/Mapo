#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_float2double (int n, double *output, float *input)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x + 
             (blockIdx.y * blockDim.x * gridDim.x) + 
             (threadIdx.y * gridDim.x * blockDim.x);

    // Check bounds and perform conversion
    if (id < n)
    {
        output[id] = (double)input[id];
    }
}