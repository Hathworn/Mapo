#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access pattern by calculating the global thread index more efficiently
__global__ void vec_scalarAdd(int n, double *result, double x, double *y)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate linear thread index
    if (id < n) 
    {
        result[id] = x + y[id];
    }
}