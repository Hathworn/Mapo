#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_divScalarMany(int n, int sizeSubImage, double *result, double *x, double *div) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    int id2 = id / sizeSubImage;  // Calculate the appropriate index for the div array
    if (id < n) {
        double divisor = div[id2];  // Load divisor once to avoid multiple indexing
        if (divisor != 0) {
            result[id] = x[id] / divisor;  // Perform division only if divisor is non-zero
        }
    }
}