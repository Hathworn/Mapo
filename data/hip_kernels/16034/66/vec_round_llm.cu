#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_round(int n, double *result, double *x) {
    // Optimize thread and block index calculation
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Avoid redundant calculations of idy and idx
    if (id < n) {
        result[id] = round(x[id]);
    }
}