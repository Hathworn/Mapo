#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//WARNING : device_sum size should be gridDim.x
__global__ void vec_sinh (int n, double *result, double *x)
{
    // Calculate global thread id
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n)
    {
        // Perform sinh operation
        result[id] = sinh(x[id]);
    }
}