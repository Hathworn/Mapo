#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_atanh (int n, double *result, double *x) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D indexing for simplicity
    if (id < n) {
        result[id] = atanh(x[id]); // Parallel computation using efficient memory access
    }
}