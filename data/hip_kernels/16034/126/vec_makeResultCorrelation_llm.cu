#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_makeResultCorrelation(int n, int sizeInput, int sizeFullPadded,float *output, float *input, int *sparse) {
    // Calculate 1D global thread index
    int id = blockIdx.y * gridDim.x * blockDim.x * blockDim.y + blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    
    if (id < n) {
        int id2 = id % sizeInput;
        int id3 = id / sizeInput;
        
        // Perform calculation
        output[id] = input[sparse[id2] + sizeFullPadded * id3] / sqrtf(sizeFullPadded / 2.0f);
    }
}