#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        // Use fast log function
        labelLogProbs[tx] = __logf(labelp);

        // Compute the probability of correct case based on max probability match
        if (labelp != maxp) {
            correctProbs[tx] = 0.0f;
        } else {
            int numMax = 0;
            // Use unrolling to potentially improve loop performance
            for (int i = 0; i < numOut; i += 4) {
                numMax += (probs[i * numCases + tx] == maxp);
                if (i + 1 < numOut) numMax += (probs[(i + 1) * numCases + tx] == maxp);
                if (i + 2 < numOut) numMax += (probs[(i + 2) * numCases + tx] == maxp);
                if (i + 3 < numOut) numMax += (probs[(i + 3) * numCases + tx] == maxp);
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}