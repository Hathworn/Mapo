#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    const uint numEls = tgtWidth * tgtHeight; // Pre-calculate total number of elements for clarity

    for (uint i = idx; i < numEls; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        tgt[i] = src[srcY * srcWidth + srcX];  // Optimal memory access pattern
    }
}