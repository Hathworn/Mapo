#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate global thread ID for better efficiency and clarity
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    // Initialize the random number generator with improved indexing
    hiprand_init(seed, tidx, 0, &state[tidx]);
}