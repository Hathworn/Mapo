#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* __restrict__ row_sum, int32_t* __restrict__ output, int32_t N) {
    int32_t i = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global index
    int32_t offset = blockIdx.x * N; // Pre-calculate offset for each block

    // Use grid-stride loop to cover all elements
    while (i < N) {
        output[offset + i] = -row_sum[blockIdx.x]; // Use pre-calculated offset
        i += blockDim.x * gridDim.x; // Advance in grid-stride
    }
}