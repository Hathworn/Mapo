#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
    int32_t index = blockIdx.x * N + threadIdx.x; // Calculate unique index for each thread
    for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
        output[index + i] = K_A_B - row_sum[blockIdx.x] - col_sum[i];
    }
}