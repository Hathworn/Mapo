#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Compute the unique thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialize random state for each thread uniquely
    hiprand_init(seed, index, 0, &state[index]);
}