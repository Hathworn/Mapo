#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Optimize by calculating the global index once
    int32_t idx = blockIdx.x * N + threadIdx.x;
    
    // Use a while loop to process all necessary elements
    while (threadIdx.x < N) {
        output[idx] = -col_sum[threadIdx.x];
        threadIdx.x += blockDim.x; // Move to the next element
        idx += blockDim.x; // Update the global index accordingly
    }
}