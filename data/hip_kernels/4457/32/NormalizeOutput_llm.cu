#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure only valid threads perform work
    if (idx < num_elements) {
        // Access global memory coalescing three elements at once
        int64_t* output = &to_normalize[idx * 3];
        output[0] = batch_index;
        output[1] = class_index;
        output[2] = static_cast<int64_t>(original[idx]);
    }
}