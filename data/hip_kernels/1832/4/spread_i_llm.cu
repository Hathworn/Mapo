#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void spread_i(Real* src, unsigned int spitch, Real* dst, unsigned int dpitch)
{
    unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x; // Optimize by storing gridDim.x in a variable
    unsigned int tid = threadIdx.x;

    if(tid < dpitch) // Move condition check outside to avoid unnecessary reads
    {
        Real res = src[bid * dpitch + tid]; // Use shared memory to coalesce memory access
        dst[bid * spitch + tid] = res; // Direct assignment without condition
    }
}