#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// Global kernel function to spread elements in y-direction with optimization
__global__ void spread_y_r(Real* src, Real* dst)
{
    // Calculate the unique thread index across the grid
    unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;
    unsigned int tid1 = tid + blockIdx.y * gridDim.x * blockDim.x;

    // Load source value into register for faster access
    Real res = src[tid];

    // Write source value to destination with an offset
    dst[tid1 + blockDim.x * gridDim.x] = res;

    // Set appropriate type of zero to other destination index
    dst[tid1] = (Real)0;
}