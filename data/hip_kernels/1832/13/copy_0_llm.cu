#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void copy_0(int nx, int ny, int nz, Real *in, Real *out) {
    // Calculate thread's unique index in a flattened grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate new positions based on grid and block indices
    int jj = (blockIdx.y * nx * ny / 4 + blockIdx.x * nx / 2) + idx;
    int jj1 = ((blockIdx.y + nz / 2) * nx * ny + (blockIdx.x + ny / 2) * nx) + idx;

    // Copy data from input to output
    out[jj1] = in[jj];
}