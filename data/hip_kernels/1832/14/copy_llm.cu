#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// Optimize thread indexing by using a single index calculation
__global__ void copy(int nx, int ny, int nz, Real *in, Real *out) {
    int tj = threadIdx.x;
    int jj = blockIdx.y * nx * ny / 4 + blockIdx.x * nx / 2 + tj;
    int jj1 = blockIdx.y * nx * ny + blockIdx.x * nx + tj;
    
    out[jj] = in[jj1];
}