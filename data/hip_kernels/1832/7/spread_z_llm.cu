#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void spread_z(Real* src, Real* dst)
{
    unsigned int tid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    // Utilize shared memory to reduce global memory access latency
    __shared__ Real tile[TILE_DIM];

    // Load data into shared memory
    tile[threadIdx.x] = src[tid];
    __syncthreads();

    // Write data to global memory from shared memory
    src[tid + (gridDim.y * gridDim.x) * blockDim.x] = tile[threadIdx.x];
    
    // Set the current location in source to zero
    #ifdef DOUBLE
    src[tid] = 0.0;
    #else
    src[tid] = 0.f;
    #endif
}