#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void spread(Real* src, unsigned int spitch, Real* dst, unsigned int dpitch)
{
    unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int tid = threadIdx.x;

    // Use shared memory for better memory access patterns
    __shared__ Real sharedData[TILE_DIM];
    
    unsigned int index = bid * spitch + tid;
    if(tid < spitch) {
        sharedData[tid] = src[index];
    }
    __syncthreads();
    
    Real res = (tid >= spitch) ? sharedData[tid-spitch] : 0.0;
    if(tid < dpitch) {
        dst[bid * dpitch + tid] = res;
    }
}