#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DOUBLE

#ifdef DOUBLE
#define Complex  hipfftDoubleComplex
#define Real double
#define Transform HIPFFT_Z2Z
#define TransformExec hipfftExecZ2Z
#else
#define Complex  hipfftComplex
#define Real float
#define Transform HIPFFT_C2C
#define TransformExec hipfftExecC2C
#endif

#define TILE_DIM  8

// synchronize blocks
__global__ void spread_y_i_r(Real* src, Real* dst)
{
    // Compute unique thread ID
    unsigned int total_threads_per_row = gridDim.x * blockDim.x;
    unsigned int row_offset = blockIdx.y * total_threads_per_row;
    unsigned int tid = row_offset + blockIdx.x * blockDim.x + threadIdx.x;

    unsigned int src_tid = tid * 2; // use stride for src as per original logic

    // Load and store
    Real res = src[src_tid];
    dst[tid] = res;
}