#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"



hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);


__global__ void addKernel(int *c, const int *a, const int *b)
{
int i = threadIdx.x;
c[i] = a[i] + b[i];
c[i] = a[i] - b[i];
}