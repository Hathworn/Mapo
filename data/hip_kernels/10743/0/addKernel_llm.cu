#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Use blockIdx.x and blockDim.x to allow kernel to process more than blockDim.x elements
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform both addition and subtraction
    int sum = a[i] + b[i];
    int diff = a[i] - b[i];
    
    // Store results in output array
    c[i] = sum;   // Optional: Modify according the logic needed (sum or diff)
    c[i] = diff;  // Remove if sum is needed instead
}