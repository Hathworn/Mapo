#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _dev_saxpy(int n, float a, float* x, float* y, float* result) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index does not exceed array bounds
    if (idx < n) {
        // Perform the SAXPY operation: result = a * x + y
        result[idx] = a * x[idx] + y[idx];
    }
}