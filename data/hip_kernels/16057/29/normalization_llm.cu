#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalization(int *glcm, float *norm, int Max, int sum) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * Max + ix;

    // Utilize shared memory to improve performance
    __shared__ int localGlcm;
    __shared__ float localSum;

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        localSum = float(sum);
    }

    if (idx < (Max + 1) * (Max + 1)) {
        localGlcm = glcm[idx];
        __syncthreads(); // Synchronize threads after updating shared memory

        // Use shared memory for normalization
        norm[idx] = float(localGlcm) / localSum;
    }
}