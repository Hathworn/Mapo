#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Div315(int *matrix, int *newMatrix, int nx, int ny, int Max){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= nx - 1 || iy >= ny) return;  // Early return if out of bounds

    int index = iy * nx + ix;
    int posisi = 0;

    // Combine conditions and iterate only over valid indices
    if (ix < nx - 1) {
        posisi = matrix[index] * nx + matrix[index + nx + 1];
        atomicAdd(&newMatrix[posisi], 1);
        // printf may be commented out in performance-sensitive code
        // printf("Index : %d %d\n", index, index + nx + 1);
    }
}