#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_IMC(float *norm, float *IMC, float *HX, float *HY, float *entropy, float *px, float *py, float *HXY, int max, float sum, int size) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * max + ix;
    int tid = threadIdx.x;

    if (idx < size) {
        float norm_value = norm[idx];
        if (norm_value > 0) {
            float log_value = log10f(norm_value);
            entropy[idx] = -norm_value * log_value;

            int row = idx / max;
            HX[idx] = -norm_value * log_value;

            atomicAdd(&px[row], norm_value);
            atomicAdd(&py[ix], norm_value);

            __syncthreads();

            if (tid == 0) {
                for (int i = 1; i < max; ++i) {
                    hx_val += HX[row * max + i];
                    hy_val += HY[i * max + ix];
                }
                HX[idx] = hx_val;
                HY[idx] = hy_val;
            }
        }
    }

    __syncthreads();

    if (idx > 9000 && norm[idx] > 0) {
        HXY[idx] = fabsf(norm[idx] * log10f(px[0] * py[0]));
    }
    
    for (int stride = 1; stride < size; stride *= 2) {
        if ((tid % (2 * stride)) == 0 && (idx + stride) < size) {
            HXY[idx] += HXY[idx + stride];
        }
        __syncthreads();
    }

    if (idx == 0) {
        if (HX[0] > HY[0]) {
            IMC[0] = (entropy[0] - HXY[0]) / HX[0];
        } else {
            IMC[0] = (entropy[0] - HXY[0]) / HY[0];
        }
        printf("IMC %f\n", fabsf(IMC[0]));
    }
}