#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Div45(int *matrix , int *newMatrix, int nx, int ny, int Max) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (ix >= nx - 1 || iy >= ny - 1) return; // Boundary check

    int Index = iy * nx + ix;
    int posisi = 0;

    // Optimize the loop by eliminating redundant calculations
    int nxTimesIndex = ix + nx * (iy + 1);

    for (int i = 0; i < nx - 1; i++) {
        if (Index >= i * nx && Index < ((i + 1) * nx) - 1) {
            posisi = matrix[nxTimesIndex - nx + 1];
            atomicAdd(&newMatrix[posisi], 1);
            printf("Index : %d %d\n", nxTimesIndex - nx, Index + 1);
        }
    }
}