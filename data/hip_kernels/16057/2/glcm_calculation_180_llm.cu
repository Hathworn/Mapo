#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void glcm_calculation_180(int *A, int *glcm, const int nx, const int ny, int max) {
    // Use threadIdx.x and blockIdx.x to compute the unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = gridDim.x * blockDim.x;

    // Iterate over the entire array with stride equal to total number of threads
    for (int i = idx; i < (nx * ny) - 1; i += totalThreads) {
        if ((i + 1) % nx != 0) { // Ensure we're not crossing row boundary
            int k = max * A[i + 1] + A[i];
            atomicAdd(&glcm[k], 1); // Use atomic operation for thread safety
        }
    }
}