#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Div90(int *matrix, int *newMatrix, int nx, int ny, int Max) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate the flattened index
    int Index = iy * nx + ix;

    // Ensure within bounds before proceeding
    if (ix < nx - 1 && iy < ny - 1 && (Index == 0 || Index % 2 == 0)) {
        // Calculate positions
        int posisi1 = matrix[Index + nx] * nx + matrix[Index];
        int posisi2 = matrix[Index + (nx + 1)] * nx + matrix[Index + 1];

        // Perform atomic adds
        atomicAdd(&newMatrix[posisi1], 1);
        atomicAdd(&newMatrix[posisi2], 1);

        // Debugging print statement (can be removed in production)
        printf("Index : %d %d dan %d %d\n", Index + nx, Index, Index + (nx + 1), Index + 1);
    }
}