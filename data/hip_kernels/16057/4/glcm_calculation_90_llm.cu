#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void glcm_calculation_90(int *A, int *glcm, const int nx, const int ny, int max) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    // Check if within bounds
    if (ix >= nx || iy >= ny - 1) return;

    unsigned int idx = iy * nx + ix;
    int k = max * A[idx + nx] + A[idx];
    // Use atomic add for parallel updates
    atomicAdd(&glcm[k], 1);
}