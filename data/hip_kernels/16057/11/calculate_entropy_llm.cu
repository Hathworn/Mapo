#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_entropy(float *norm, float *entropy, int max, float sum, int size) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Initialize shared memory
    extern __shared__ float shared_entropy[];

    float local_entropy = 0.0f;
    
    // Calculate entropy contribution for each thread's portion of the array
    while (idx < size) {
        if (norm[idx] != 0) {
            local_entropy -= norm[idx] * log10f(norm[idx]);
        }
        idx += stride;
    }

    // Store each thread's partial result into shared memory
    int tid = threadIdx.x;
    shared_entropy[tid] = local_entropy;
    __syncthreads();

    // Perform reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_entropy[tid] += shared_entropy[tid + s];
        }
        __syncthreads();
    }

    // Add the block's result to the global memory
    if (tid == 0) {
        atomicAdd(&entropy[0], shared_entropy[0]);
    }

    // Print final entropy result from one thread
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        printf("entropy %f\n", entropy[0]);
    }
}