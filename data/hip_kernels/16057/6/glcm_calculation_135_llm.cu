#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void glcm_calculation_135(int *A, int *glcm, const int nx, const int ny, int max) {
    // Compute global thread IDs
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Calculate unique index for the 1D array
    unsigned int idx = iy * nx + ix;
    
    // Check if within bounds
    if (ix >= 0 && ix < nx - 1 && iy >= 0 && iy < ny - 1) {
        int k = max * A[idx] + A[idx + nx - 1]; // 135-degree offset
        atomicAdd(&glcm[k], 1); // Atomic add for safe parallel updates
    }
    __syncthreads(); // Synchronize all threads
}