#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void glcm_calculation_45(int *A, int *glcm, const int nx, const int ny, int max) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    
    // Iterate only within valid indices and grid boundaries
    if (ix > 0 && ix < nx && iy < ny) {  // Ensure threads process valid elements
        int currentBlock = blockIdx.x;
        if (idx < nx * (currentBlock + 1) - 1) {  // Check within block valid range
            int k = max * A[idx] + A[idx - (nx - 1)];
            atomicAdd(&glcm[k], 1);  // Safely update glcm with atomic operation
        }
    }
}