#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void glcm_calculation_315(int *A, int *glcm, const int nx, const int ny, int max) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx - 1 && iy < ny - 1) { // Limit access to valid indices
        int k = max * A[idx] + A[idx + nx + 1]; // Calculate index for glcm
        atomicAdd(&glcm[k], 1); // Atomic addition to prevent race conditions
    }
}