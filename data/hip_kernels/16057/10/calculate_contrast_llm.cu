#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_contrast(float *norm, float *contrast, int *dif, int max, float sum, int size) {
    // Calculate global index
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * max + ix;
    int tid = threadIdx.x;

    // Return if index is out of bounds
    if (idx >= max * max) return;

    // Perform element-wise multiplication and store results in contrast array
    if (idx < size) {
        contrast[idx] = norm[idx] * dif[idx];
    }

    // Perform in-place reduction within each block
    for (int stride = 1; stride < max; stride *= 2) {
        // Ensure active threads only
        if ((tid % (2 * stride)) == 0 && (idx + stride) < size) {
            contrast[idx] += contrast[idx + stride];
        }
        __syncthreads();  // Synchronize within threadblock
    }

    // Print result at the first element of the array
    if (idx == 0) {
        printf("Contrast: %f\n", contrast[0]);
    }
}