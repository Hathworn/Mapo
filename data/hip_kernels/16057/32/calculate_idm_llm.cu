#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_idm(float *norm, float *idm, int *dif, int max, float sum, int size) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * max + ix;
    int tid = threadIdx.x;

    if (idx < size) {
        // Compute intermediate IDM value
        idm[idx] = ((float(1) / (1 + dif[idx])) * norm[idx]);
    }
    __syncthreads(); // Ensure all threads have computed idm[idx]

    // Optimized reduction with loop unrolling
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride && (idx + stride) < size) {
            idm[idx] += idm[idx + stride];
        }
        __syncthreads();
    }

    // Print result from the first thread of the first block
    if (idx == 0) {
        printf("idm %f\n", idm[0]);
    }
}