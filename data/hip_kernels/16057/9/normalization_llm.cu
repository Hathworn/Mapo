#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalization(int *glcm, float *norm, int max, int sum) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * max + ix;
    
    // Ensure threads sync before any operation
    __syncthreads();
    
    // Efficient boundary check
    if (ix < max && iy < max) {
        norm[idx] = static_cast<float>(glcm[idx]) / sum;  // Use static_cast for clarity
    }
}