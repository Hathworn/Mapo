#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void glcm_calculation_225(int *A, int *glcm, const int nx, const int ny, int max) {
    // Compute global thread index based on block and thread indices
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;
    
    // Check if the thread index is within bounds
    if (ix >= nx || iy >= ny) return;

    // Optimize: Loop unrolling and conditional check before entering atomicAdd
    int k = 0;
    if (blockIdx.x >= 1 && idx > blockIdx.x * nx) {
        k = max * A[idx] + A[idx - (nx + 1)];
        // Atomic operation to avoid race conditions
        atomicAdd(&glcm[k], 1);
    }

    __syncthreads(); // Synchronize threads within a block
}