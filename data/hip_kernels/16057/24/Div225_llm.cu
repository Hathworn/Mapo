#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Div225(int *matrix, int *newMatrix, int nx, int ny, int Max) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int Index = iy * nx + ix;
    
    // Check if within bounds
    if (ix < nx - 1 && iy < ny - 1) {
        int posisi = matrix[Index + 1] * nx + matrix[Index + nx];
        // Atomic add to prevent race conditions
        atomicAdd(&newMatrix[posisi], 1);
    }
}