#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Div0(int *matrix, int *newMatrix, int nx, int ny, int Max) {
    // Calculate global thread index.
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int Index = iy * nx + ix;
    
    // Check boundary conditions for valid index.
    if (ix < nx - 1 && iy < ny - 1) { 
        for (int i = 0; i < nx; i += 2) {
            if (Index >= i * nx && Index < ((i + 1) * nx) - 1) {
                // Precompute matrix values once to reduce redundant calculations.
                int matrix_val = matrix[Index];
                int matrix_val_plus1 = matrix[Index + 1];
                int matrix_val_plusMax = matrix[Index + Max];
                int matrix_val_plusMax1 = matrix[Index + (Max + 1)];

                // Calculate position and perform atomic adds.
                int posisi1 = matrix_val * Max + matrix_val_plus1;
                atomicAdd(&newMatrix[posisi1], 1);

                int posisi2 = matrix_val_plusMax * Max + matrix_val_plusMax1;
                atomicAdd(&newMatrix[posisi2], 1);
            }
        }
    }
}