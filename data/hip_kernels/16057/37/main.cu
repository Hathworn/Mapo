#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "calculate_IMC.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *norm = NULL;
hipMalloc(&norm, XSIZE*YSIZE);
float *IMC = NULL;
hipMalloc(&IMC, XSIZE*YSIZE);
float *HX = NULL;
hipMalloc(&HX, XSIZE*YSIZE);
float *HY = NULL;
hipMalloc(&HY, XSIZE*YSIZE);
float *entropy = NULL;
hipMalloc(&entropy, XSIZE*YSIZE);
float *px = NULL;
hipMalloc(&px, XSIZE*YSIZE);
float *py = NULL;
hipMalloc(&py, XSIZE*YSIZE);
float *HXY = NULL;
hipMalloc(&HXY, XSIZE*YSIZE);
int max = 1;
float sum = 1;
int size = XSIZE*YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
calculate_IMC<<<gridBlock, threadBlock>>>(norm,IMC,HX,HY,entropy,px,py,HXY,max,sum,size);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
calculate_IMC<<<gridBlock, threadBlock>>>(norm,IMC,HX,HY,entropy,px,py,HXY,max,sum,size);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
calculate_IMC<<<gridBlock, threadBlock>>>(norm,IMC,HX,HY,entropy,px,py,HXY,max,sum,size);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}