#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void calculate_IMC(float *norm,float *IMC,float *HX,float *HY,float *entropy,float *px,float *py,float *HXY,int max,float sum,int size) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * max + ix;
    int tid = threadIdx.x;

    // Calculate HX if conditions are met
    if (idx < max * max && norm[idx] > 0) {
        int row = idx / max;
        if (idx < (row + 1) * max) {
            HX[idx] = -(norm[idx] * log10f(norm[idx]));
        }
    }

    // Calculate entropy if conditions are met
    if (idx < size && norm[idx] != 0) {
        entropy[idx] = -(norm[idx] * log10f(norm[idx]));
    }
    __syncthreads();

    // Assign px value
    if (idx < size) {
        px[idx] = norm[idx];
    }

    // Calculate py if conditions are met
    if (idx < max) {
        for (int i = 0; i < max; ++i) {
            py[i * max + idx] = norm[idx * max + i];
        }
    }

    // Calculate HY if conditions are met
    if (idx < max) {
        for (int i = 0; i < max; ++i) {
            if (norm[idx * max + i] > 0) {
                HY[i * max + idx] = -(norm[idx * max + i] * log10f(norm[idx * max + i]));
            }
        }
    }

    // Use shared memory for reduction optimization
    __shared__ float sharedHX[MAX_BLOCK_SIZE];
    __shared__ float sharedHY[MAX_BLOCK_SIZE];
    __shared__ float sharedPx[MAX_BLOCK_SIZE];
    __shared__ float sharedPy[MAX_BLOCK_SIZE];
    __shared__ float sharedEntropy[MAX_BLOCK_SIZE];

    sharedHX[tid] = HX[idx];
    sharedHY[tid] = HY[idx];
    sharedPx[tid] = px[idx];
    sharedPy[tid] = py[idx];
    sharedEntropy[tid] = entropy[idx];
    __syncthreads();

    // Reduction over thread block
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if (tid % (2 * stride) == 0) {
            sharedHX[tid] += sharedHX[tid + stride];
            sharedHY[tid] += sharedHY[tid + stride];
            sharedPx[tid] += sharedPx[tid + stride];
            sharedPy[tid] += sharedPy[tid + stride];
            sharedEntropy[tid] += sharedEntropy[tid + stride];
        }
        __syncthreads();
    }

    // Write results from shared memory to global memory
    if (tid == 0) {
        atomicAdd(&HX[0], sharedHX[0]);
        atomicAdd(&HY[0], sharedHY[0]);
        atomicAdd(&px[0], sharedPx[0]);
        atomicAdd(&py[0], sharedPy[0]);
        atomicAdd(&entropy[0], sharedEntropy[0]);
    }

    __syncthreads();

    // Calculate HXY and perform reduction
    if (idx > 9000) {
        HXY[idx] = fabsf(norm[idx] * log10f(px[0] * py[0]));
    }
    __syncthreads();

    // Use shared memory for HXY reduction
    __shared__ float sharedHXY[MAX_BLOCK_SIZE];
    sharedHXY[tid] = HXY[idx];
    __syncthreads();

    // Reduction over thread block for HXY
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if (tid % (2 * stride) == 0) {
            sharedHXY[tid] += sharedHXY[tid + stride];
        }
        __syncthreads();
    }

    // Write result from shared memory to global memory for HXY
    if (tid == 0) {
        atomicAdd(&HXY[0], sharedHXY[0]);
    }

    __syncthreads();

    // Calculate final IMC at index 0
    if (idx == 0) {
        if (HX[0] > HY[0]) {
            IMC[0] = (entropy[0] - HXY[0]) / HX[0];
        } else {
            IMC[0] = (entropy[0] - HXY[0]) / HY[0];
        }
        printf("IMC %f\n", fabsf(IMC[0]));
    }
}