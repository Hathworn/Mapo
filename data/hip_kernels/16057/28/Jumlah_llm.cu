#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Jumlah(float *sumMatrix, float *mulMatrix) {
    int Index = blockIdx.x * blockDim.x + threadIdx.x;

    // Load value from global memory to register for faster access
    float value = 0.0f;
    if (Index < 1) {
        value = mulMatrix[0];
        printf("%f", value);
    } else {
        value = mulMatrix[Index];
    }

    // Use atomic add to prevent race conditions while updating sumMatrix
    atomicAdd(&sumMatrix[0], value);
}