#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Div180(int *matrix, int *newMatrix, int nx, int ny, int Max) {
    // Compute the global index
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Compute the linear index
    int Index = iy * nx + ix;

    // Check if the index is within the valid range before processing
    if (ix < nx && iy < ny) {
        int posisi = 0;
        
        // Ensure that the loops do not exceed array bounds
        for (int i = 0; i < nx; i += 2) {
            if (Index >= i * nx && Index < ((i + 1) * nx) - 1) {
                
                // First atomic addition
                posisi = matrix[Index + 1] * nx + matrix[Index];
                atomicAdd(&newMatrix[posisi], 1);
                
                // Second atomic addition
                // Add a boundary check to prevent out-of-bounds access
                if(Index + (nx + 1) < nx * ny) {
                    posisi = matrix[Index + (nx + 1)] * nx + matrix[Index + nx];
                    atomicAdd(&newMatrix[posisi], 1);
                }
            }
        }
    }
}