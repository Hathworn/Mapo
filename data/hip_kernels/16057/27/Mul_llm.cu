#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Mul(float *newMatrix, float *mulMatrix, int Max, float *sumMatrix) {
    // Use shared memory to reduce global memory accesses
    __shared__ float sharedNewMatrix[32][32]; // Adjust size as needed
    __shared__ float sharedMulMatrix[32][32]; // Adjust size as needed

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float temp = 0.0f;
    for (int k = 0; k < Max; k++) {
        // Load elements into shared memory
        sharedNewMatrix[threadIdx.y][threadIdx.x] = newMatrix[row * Max + k];
        sharedMulMatrix[threadIdx.y][threadIdx.x] = newMatrix[k * Max + col];
        __syncthreads(); // Synchronize threads for shared memory use

        // Calculate product and accumulate into temp
        temp += sharedNewMatrix[threadIdx.y][threadIdx.x] * sharedMulMatrix[threadIdx.y][threadIdx.x];
        __syncthreads(); // Ensure all computations are complete before next iteration
    }

    // Write results to global memory
    atomicAdd(&mulMatrix[row * Max + col], temp);
}