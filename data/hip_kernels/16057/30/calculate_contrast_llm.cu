#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_contrast(float *norm, float *contrast, int *dif, int max, float sum, int size) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * max + ix;

    // Boundary check to ensure idx is within limits
    if (idx >= size) return;

    contrast[idx] = norm[idx] * dif[idx];

    // In-block reduction
    for (int stride = 1; stride < blockDim.x; stride *= 2) { 
        __syncthreads(); // Synchronize before accessing shared data
        if (threadIdx.x % (2 * stride) == 0 && idx + stride < size) {
            contrast[idx] += contrast[idx + stride];
        }
    }

    // Print the result from the first thread of the first block
    if (idx == 0) {
        printf("contrast %f\n", contrast[0]);
    }
}