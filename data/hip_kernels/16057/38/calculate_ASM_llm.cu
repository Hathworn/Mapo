#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_ASM(float *norm, float *ASM, float *mulMatrix, int Max) {
    // Calculate global row and column index
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if we are within the bounds
    if (row < Max && col < Max) {
        
        // Accumulate results for a single element using shared memory
        float sum = 0.0f;
        for (int k = 0; k < Max; k++) {
            sum += norm[row * Max + k] * norm[k * Max + col];
        }
        
        // Store the result atomically
        atomicAdd(&mulMatrix[row * Max + col], sum);
    }
    
    // Use a single thread to accumulate ASM
    if (col == 0 && row == 0) {
        float asmSum = 0.0f;
        // Atomic addition over ASM array
        for (int i = 0; i < Max * Max; i++) {
            atomicAdd(&asmSum, mulMatrix[i]);
        }
        atomicAdd(&ASM[0], asmSum);
        printf("ASM %f\n", ASM[0]);
    }
}