#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "calculate_correlation.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *norm = NULL;
hipMalloc(&norm, XSIZE*YSIZE);
float *corelation = NULL;
hipMalloc(&corelation, XSIZE*YSIZE);
float *miu_x = NULL;
hipMalloc(&miu_x, XSIZE*YSIZE);
float *miu_y = NULL;
hipMalloc(&miu_y, XSIZE*YSIZE);
float *stdx = NULL;
hipMalloc(&stdx, XSIZE*YSIZE);
float *stdy = NULL;
hipMalloc(&stdy, XSIZE*YSIZE);
int *ikj = NULL;
hipMalloc(&ikj, XSIZE*YSIZE);
float *dif_variance = NULL;
hipMalloc(&dif_variance, XSIZE*YSIZE);
int max = 1;
float sum = 1;
int size = XSIZE*YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
calculate_correlation<<<gridBlock, threadBlock>>>(norm,corelation,miu_x,miu_y,stdx,stdy,ikj,dif_variance,max,sum,size);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
calculate_correlation<<<gridBlock, threadBlock>>>(norm,corelation,miu_x,miu_y,stdx,stdy,ikj,dif_variance,max,sum,size);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
calculate_correlation<<<gridBlock, threadBlock>>>(norm,corelation,miu_x,miu_y,stdx,stdy,ikj,dif_variance,max,sum,size);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}