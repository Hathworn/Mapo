#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_correlation(float *norm, float *corelation, float *miu_x, float *miu_y, float *stdx, float *stdy, int *ikj, float *dif_variance, int max, float sum, int size) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * max + ix;
    int tid = threadIdx.x;

    // Combined and simplified loops for miu_x and stdx computation
    for (int i = 0; i < max; i++) {
        if (idx >= i * max && idx < (i + 1) * max) {
            miu_x[idx] = i * norm[idx];
            stdx[idx] = ((i - miu_x[0]) * (i - miu_x[0])) * norm[idx];
        }
    }

    // Combined and simplified loops for miu_y and stdy computation
    for (int i = 0, blok = 0; i < max; i++) {
        if (blok == i && idx < max) {
            miu_y[blok * max + idx] = i * norm[idx * max + i];
            stdy[blok * max + idx] = ((i - miu_y[0]) * (i - miu_y[0])) * norm[idx * max + i];
            blok++;
        }
    }

    // Initialization of ikj matrix
    if (idx == 0) {
        for (int i = 0; i < max; i++) {
            for (int j = 0; j < max; j++) {
                ikj[max * i + j] = i * j;
            }
        }
    }

    // Compute corelation
    if (idx < size) {
        corelation[idx] = (ikj[idx] * norm[idx]);
    }

    // Parallel reduction for corelation
    for (int stride = 1; stride < size; stride *= 2) {
        if ((tid % (2 * stride)) == 0) {
            corelation[idx] += corelation[idx + stride];
        }
        __syncthreads();
    }

    // Parallel reduction for miu and stdy, stdx
    for (int stride = 1; stride < size; stride *= 2) {
        if ((tid % (2 * stride)) == 0) {
            miu_x[idx] += miu_x[idx + stride];
            stdy[idx] += stdy[idx + stride];
            miu_y[idx] += miu_y[idx + stride];
            stdx[idx] += stdx[idx + stride];
        }
        __syncthreads();
    }

    // Compute dif_variance
    if (idx == 0) {
        for (int i = 0; i < max; i++) {
            for (int j = 0; j < max; j++) {
                int k = abs(i - j);
                dif_variance[k] = ((k - ((miu_x[0] + miu_y[0]) / 2)) * (k - ((miu_x[0] + miu_y[0]) / 2))) * norm[k];
                if (k == i) {
                    dif_variance[k] += dif_variance[i];
                }
            }
        }
    }

    // Parallel reduction for dif_variance
    for (int stride = 1; stride < size; stride *= 2) {
        if ((tid % (2 * stride)) == 0) {
            dif_variance[idx] += dif_variance[idx + stride];
        }
        __syncthreads();
    }

    // Print results
    if (idx == 0) {
        printf("correlation %f\n", abs(corelation[0] - miu_x[0] * miu_y[0]) / (stdx[0] * stdy[0]));
        printf("variance %f\n", stdx[0]);
        printf("difference variance %f\n", dif_variance[0]);
    }
}