#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_ASM(float *norm, float *ASM, int max, float sum, int size) {
    // Calculate global thread indices for 2D grid and block
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * max + ix;
    int tid = threadIdx.x;

    // Check if index is within bounds
    if (idx < size) {
        // Compute ASM value for the current index
        ASM[idx] = norm[idx] * norm[idx];
    }

    // Perform parallel reduction to compute sum of ASMs
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        // Ensure the first thread in stride can sum with its neighbor
        if (tid < stride && idx + stride < size) {
            ASM[idx] += ASM[idx + stride];
        }
        // Synchronize threads to ensure all computations are complete before next iteration
        __syncthreads();
    }

    // Print the final result from the first thread of the block
    if (tid == 0 && idx < size) {
        printf("ASM %f %d\n", ASM[0], idx);
    }
}