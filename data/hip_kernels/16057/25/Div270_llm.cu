#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Div270(int *matrix , int *newMatrix,int nx,int ny,int Max){
    // Calculate the unique index for the current thread
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    
    int Index = iy * nx + ix;

    // Ensure we are within bounds
    if (ix < nx - 1 && iy < ny - 1){

        // Check if Index is even or zero for efficient processing
        if (Index == 0 || Index % 2 == 0) {
            // Optimized calculation for 'posisi' and atomic addition to 'newMatrix'
            int posisi1 = matrix[Index] * nx + matrix[Index + nx];
            int posisi2 = matrix[Index + 1] * nx + matrix[Index + (nx + 1)];
            
            atomicAdd(&newMatrix[posisi1], 1);
            atomicAdd(&newMatrix[posisi2], 1);

            // Debug print statement
            printf("Index : %d %d dan %d %d\n", Index, Index + nx , Index + 1, Index + (nx + 1));
        }
    }
}