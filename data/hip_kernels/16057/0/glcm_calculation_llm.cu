#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void glcm_calculation(int *A, int *glcm, float *glcmNorm, const int nx, const int ny, int maxx)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    // Calculate GLCM
    if (idx < nx * ny) {
        int k, l;
        for (k = 0; k <= maxx; k++) {
            for (l = 0; l <= maxx; l++) {
                if (A[idx] == k && A[idx + 1] == l) {
                    int p = (maxx + 1) * k + l;
                    atomicAdd(&glcm[p], 1); // Use atomicAdd to prevent race conditions
                }
            }
        }
    }

    // Normalization
    if (idx < (maxx + 1) * (maxx + 1)) {
        int sum = 0;
        __shared__ int shared_sum; // Use shared memory to accelerate sum calculation

        if (threadIdx.x == 0 && threadIdx.y == 0) {
            for (int k = 0; k < (maxx + 1) * (maxx + 1); k++) {
                sum += glcm[k];
            }
            shared_sum = sum;
        }
        __syncthreads();

        glcmNorm[idx] = float(glcm[idx]) / float(shared_sum);
    }

    // Additional Calculations
    if (ix < 1) {
        float sums = 0;
        float f1 = 0;
        float f2 = 0;
        float f4 = 0;

        for (int k = 0; k < (maxx + 1) * (maxx + 1); k++) {
            sums += glcmNorm[k];
            f1 += glcmNorm[k];
            f2 += k * k * sums;
            if (glcmNorm[k] > 0) f4 += glcmNorm[k] * log10f(glcmNorm[k]);
        }

        float f3 = sqrt(f1);

        printf("ASM : %.1f\n", f1);
        printf("Contrast : %.1f\n", f2);
        printf("Energy : %.1f\n", f3);
        printf("Entropy : %.1f\n", f4);
    }
}