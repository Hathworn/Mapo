#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Div135(int *matrix , int *newMatrix,int nx,int ny,int Max){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    int Index = iy * nx + ix;
    if (Index >= (nx - 1) * ny) return; // Prevent out-of-bound Index
    
    int i = Index / nx; // Calculate current row
    if (Index < (i + 1) * nx - 1) { // Check index range
        int posisi = matrix[Index + (nx + 1)] * nx + matrix[Index];
        atomicAdd(&newMatrix[posisi], 1);
        printf("Index : %d %d\n", Index + (nx + 1), Index);
    }
}