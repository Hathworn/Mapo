#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_entropy(float *norm, float *entropy, int max, float sum, int size) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * max + ix;
    int tid = threadIdx.x;

    if (idx < size && norm[idx] != 0) {
        entropy[idx] = -(norm[idx] * log10f(norm[idx]));
    }
    __syncthreads(); // Ensure all threads have computed entropy

    // Optimize reduction using stride loop
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if (tid % (2 * stride) == 0 && (idx + stride) < size) {
            entropy[idx] += entropy[idx + stride];
        }
        __syncthreads(); // Synchronize threads within the block
    }

    // Output result from the first thread of the block
    if (tid == 0) {
        printf("entropy %f\n", entropy[ix + iy * blockDim.x * gridDim.x]);
    }
}