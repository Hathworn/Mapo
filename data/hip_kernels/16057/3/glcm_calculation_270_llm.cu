#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void glcm_calculation_270(int *A, int *glcm, const int nx, const int ny, int max) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny - 1) {  // Ensure thread operates within bounds
        int k = max * A[idx] + A[idx + nx]; // Calculate the glcm index
        atomicAdd(&glcm[k], 1); // Atomic update to glcm
    }
}