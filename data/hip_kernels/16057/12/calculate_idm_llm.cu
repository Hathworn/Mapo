#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculate_idm(float *norm, float *idm, int *dif, int max, float sum, int size) {
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * max + ix;
    int tid = threadIdx.x;

    if (idx < size) {
        // Calculate individual idm values using shared memory for faster memory access
        float idm_value = ((float(1) / (1 + dif[idx])) * norm[idx]);

        __shared__ float shared_idm[1024];
        shared_idm[tid] = idm_value;
        __syncthreads();

        // Perform parallel reduction on shared memory for optimized summation
        for (int stride = 1; stride < blockDim.x; stride *= 2) {
            if ((tid % (2 * stride)) == 0 && (tid + stride) < blockDim.x) {
                shared_idm[tid] += shared_idm[tid + stride];
            }
            __syncthreads();
        }

        // Store the result back to global memory
        if (tid == 0) {
            atomicAdd(&idm[ix], shared_idm[0]);  // Ensure atomic addition to handle concurrent writes
        }
    }

    if (idx == 0) {
        printf("idm %f\n", idm[0]);
    }
}