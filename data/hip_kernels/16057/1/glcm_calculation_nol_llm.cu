#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void glcm_calculation_nol(int *A, int *glcm, const int nx, const int ny, int maxx)
{
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    // Ensure the next element exists in A to prevent out of bounds access
    if (ix < nx - 1 && iy < ny) 
    {
        int a_idx = A[idx];
        int a_idx1 = A[idx + 1];

        // Use shared memory to reduce repeated cleanup on global memory
        __shared__ int local_glcm[1024]; // This should fit maxx + 1 elements
        int threadIndex = threadIdx.y * blockDim.x + threadIdx.x;

        if (threadIndex <= maxx * maxx) 
        {
            local_glcm[threadIndex] = 0;
        }
        __syncthreads();

        if (a_idx <= maxx && a_idx1 <= maxx) 
        {
            int p = a_idx * (maxx + 1) + a_idx1;
            atomicAdd(&local_glcm[p], 1);
        }
        __syncthreads();

        if (threadIndex <= maxx * maxx) 
        {
            atomicAdd(&glcm[threadIndex], local_glcm[threadIndex]);
        }
    }
}