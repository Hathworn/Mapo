#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function using shared memory for efficient access
__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod, const size_t lod_len, size_t* out_lod0) {
    // Calculating global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (index < lod_len) {
        // Load value of in_lod[index] once into a register
        size_t lod_value = in_lod[index];
        
        // Calculate and store result using the register value
        out_lod0[index] = lod_value - num_erased[lod_value];
    }
}