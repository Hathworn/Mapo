#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Unroll the loop to improve memory access patterns
    for (int i = idx; i < dim; i += blockDim.x * 4) {
        if (i < dim) tmp += dY[i] * Y[i];
        if (i + blockDim.x < dim) tmp += dY[i + blockDim.x] * Y[i + blockDim.x];
        if (i + 2 * blockDim.x < dim) tmp += dY[i + 2 * blockDim.x] * Y[i + 2 * blockDim.x];
        if (i + 3 * blockDim.x < dim) tmp += dY[i + 3 * blockDim.x] * Y[i + 3 * blockDim.x];
    }
    
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Optimize the reduction using a tree-based approach
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    // Compute gradient
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}