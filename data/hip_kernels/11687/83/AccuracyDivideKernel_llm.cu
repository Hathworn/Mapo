#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use atomic operation to ensure correctness while dividing and updating the value
    atomicExch(accuracy, (*accuracy / N));
}