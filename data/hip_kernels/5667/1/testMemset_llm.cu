#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef __cplusplus
extern "C" {
#endif

#ifdef __cplusplus
}
#endif

__global__ void testMemset(float* array, float value, int N) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    if (i < N) {
        array[i] = value; // Assign value if within bounds
    }
}