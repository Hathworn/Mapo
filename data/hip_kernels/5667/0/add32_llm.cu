#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void add32(float* A, float *B, int size) {
    // Use blockDim.x, y, z and threadIdx.x, y, z for more concise index calculation
    int index = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (threadIdx.z + blockDim.z * (blockIdx.x + blockDim.x * (blockIdx.y + blockDim.y * blockIdx.z))));
    if (index < size) {
        A[index] += B[index];
    }
}