#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Calculate global thread index
    int32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds and process each element
    while (index < N) {
        output[blockIdx.x * N + index] = -col_sum[index];
        index += blockDim.x * gridDim.x;  // Move to the next stripe handled by this thread
    }
}