#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Use blockIdx.x to give each block a unique seed
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}