#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Calculate a unique index for each thread across all blocks
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        // Use the unique index to perform calculations directly
        output[idx] = -row_sum[blockIdx.x];
    }
}