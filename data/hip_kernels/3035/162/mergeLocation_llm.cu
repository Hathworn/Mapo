#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* __restrict__ loc_, float* __restrict__ x, float* __restrict__ y, const int npoints, float scale)
{
    // Calculate point index using blockDim, blockIdx, and threadIdx
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure that we operate only within bounds
    if (ptidx < npoints)
    {
        // Fetch loc value from global memory once
        short2 loc = loc_[ptidx];

        // Perform computations and store results in the global memory
        x[ptidx] = __fmul_rn(loc.x, scale); // Use fast multiply intrinsic
        y[ptidx] = __fmul_rn(loc.y, scale); // Use fast multiply intrinsic
    }
}