#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_transpo_kernel_naive(u_char *Source, u_char *Resultat, unsigned width, unsigned height) {
    // Calculate global column and row indices
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Check within bounds and perform operation
    if (i < height && j < width) {
        Resultat[j * height + i] = Source[i * width + j];
    }
}