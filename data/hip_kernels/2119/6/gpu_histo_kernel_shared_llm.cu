#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_histo_kernel_shared(u_char* Source, int *res, unsigned height, unsigned width) {
    __shared__ int hist[256];

    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    int index = threadIdx.y * blockDim.x + threadIdx.x;  // Correct indexing for 2D threads

    // Initialize shared histogram to zero
    if (index < 256) {
        hist[index] = 0;
    }
    __syncthreads();

    // Ensure the thread is within bounds
    if (i < height && j < width) {
        atomicAdd(&hist[Source[i * width + j]], 1);
    }
    __syncthreads();

    // Accumulate histogram to global memory
    if (index < 256) {
        atomicAdd(&res[index], hist[index]);
    }
}