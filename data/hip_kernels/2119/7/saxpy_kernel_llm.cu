#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_kernel(int n, float a, float *v1, float *v2, float *s) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for coalesced global memory access
    __shared__ float s_v1[256];

    if (i < n) {
        // Load v1 data into shared memory
        s_v1[threadIdx.x] = v1[i];
        __syncthreads();

        // Perform computation using shared memory
        s[i] = a * s_v1[threadIdx.x] + v2[i];
    }
}