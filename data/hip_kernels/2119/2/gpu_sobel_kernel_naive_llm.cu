#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_sobel_kernel_optimized(u_char *Source, u_char *Resultat, unsigned width, unsigned height) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int globalIndex = i * width + j;

    // Check bounds and avoid unnecessary computation
    if (i > 0 && i < height - 1 && j > 0 && j < width - 1) {
        int gx = Source[(i-1)*width+(j+1)] + 2*Source[i*width+(j+1)] + Source[(i+1)*width+(j+1)]
                - Source[(i-1)*width+(j-1)] - 2*Source[i*width+(j-1)] - Source[(i+1)*width+(j-1)];

        int gy = Source[(i-1)*width+(j-1)] + 2*Source[(i-1)*width+j] + Source[(i-1)*width+(j+1)]
                - Source[(i+1)*width+(j-1)] - 2*Source[(i+1)*width+j] - Source[(i+1)*width+(j+1)];

        Resultat[globalIndex] = min(255, int(sqrtf(gx * gx + gy * gy)));
    } else {
        Resultat[globalIndex] = 0;
    }
}