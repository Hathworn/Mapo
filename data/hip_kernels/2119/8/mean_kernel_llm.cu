#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function with improved memory coalescing
__global__ void mean_kernel(int n, float* v1, float* v2, float* res){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop through data with stride to improve occupancy
    for (int i = idx; i < n; i += stride) {
        res[i] = (v1[i] + v2[i]) * 0.5f; // Use multiplication for division by 2
    }
}