#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_transpo_kernel_shared(u_char *Source, u_char *Resultat, unsigned width, unsigned height) {
    __shared__ u_char tuile[BLOCKDIM_X][BLOCKDIM_Y+1];

    int x = threadIdx.x;
    int y = threadIdx.y;
    int i = blockIdx.y * BLOCKDIM_Y + y;
    int j = blockIdx.x * BLOCKDIM_X + x;

    // Check bounds to avoid unnecessary calculations
    if (i < height && j < width) {
        tuile[y][x] = Source[i * width + j];
    }
    __syncthreads();

    // Recalculate indices for efficient transpose writing
    i = blockIdx.y * BLOCKDIM_Y + x;
    j = blockIdx.x * BLOCKDIM_X + y;

    // Check bounds for transpose writing
    if (j < height && i < width) {
        Resultat[j * height + i] = tuile[x][y];
    }
}