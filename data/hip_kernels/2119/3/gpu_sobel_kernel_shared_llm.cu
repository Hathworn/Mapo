#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_sobel_kernel_shared(u_char *Source, u_char *Resultat, unsigned width, unsigned height) {
    __shared__ u_char tuile[BLOCKDIM_X][BLOCKDIM_Y];

    int x = threadIdx.x;
    int y = threadIdx.y;
    int i = blockIdx.y * (BLOCKDIM_Y-2) + y;
    int j = blockIdx.x * (BLOCKDIM_X-2) + x;

    int globalIndex = i * width + j;

    // Load pixels into shared memory
    if (i < height && j < width) {
        tuile[y][x] = Source[globalIndex];
    }
    __syncthreads();

    // Ensure threads operate within valid boundaries
    if (x > 0 && y > 0 && x < BLOCKDIM_X-1 && y < BLOCKDIM_Y-1 && i < height-1 && j < width-1) {
        u_char val_x = abs(tuile[y-1][x-1] + tuile[y-1][x] + tuile[y-1][x+1]
                         - tuile[y+1][x-1] - tuile[y+1][x] - tuile[y+1][x+1]);
        u_char val_y = abs(tuile[y-1][x-1] + tuile[y][x-1] + tuile[y+1][x-1]
                         - tuile[y-1][x+1] - tuile[y][x+1] - tuile[y+1][x+1]);
        Resultat[globalIndex] = val_x + val_y;
    }
}