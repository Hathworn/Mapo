#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized for improved performance
__global__ void gpu_dotp_kernel(int size, float* vec1, float* vec2, float* res) {
    extern __shared__ float shared_cache[]; // Use shared memory
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x; // Thread index

    shared_cache[tid] = (i < size) ? vec1[i] * vec2[i] : 0.0f; // Store partial results in shared memory
    __syncthreads(); // Synchronize threads within the block

    // Perform reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_cache[tid] += shared_cache[tid + stride];
        }
        __syncthreads(); // Ensure all additions are completed
    }

    // Store the result in global memory
    if (tid == 0) {
        atomicAdd(res, shared_cache[0]); // Add the block's result to global result
    }
}