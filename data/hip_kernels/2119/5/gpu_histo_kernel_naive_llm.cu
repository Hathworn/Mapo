#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_histo_kernel_naive(u_char* Source, int *res, unsigned height, unsigned width) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread indices are within image boundaries
    if ((i < height) && (j < width)) {
        // Load value from source image
        u_char val = Source[i * width + j];
        // Perform atomic add to the histogram result
        atomicAdd(&res[val], 1);
    }
}