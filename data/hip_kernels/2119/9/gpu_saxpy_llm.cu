#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_saxpy(int n, float a, float *x, float *y, float *s)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop to enable every thread to perform multiple operations if necessary
    for (int j = i; j < n; j += blockDim.x * gridDim.x) {
        s[j] = a * x[j] + y[j];
    }
}