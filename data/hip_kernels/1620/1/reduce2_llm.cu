#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define subCOL 5248
#define COL 5248
#define ROW 358
#define WARPABLEROW 512
#define blocksize 256
#define subMatDim subCOL*WARPABLEROW
#define targetMatDim ROW * COL

__global__ void reduce2(int *g_idata, int *g_odata, int g_size)
{
    __shared__ int sdata[blocksize];
    
    // Each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + tid;
    
    // Avoid out-of-bounds access
    sdata[tid] = (i < g_size) ? g_idata[i] : 0;
    
    __syncthreads();

    // Do reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}