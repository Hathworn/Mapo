#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define subCOL 5248
#define COL 5248
#define ROW 358
#define WARPABLEROW 512
#define blocksize 256
#define subMatDim subCOL*WARPABLEROW
#define targetMatDim ROW * COL

__global__ void reduce5(int *g_idata, int *g_odata, int g_size)
{
    __shared__ int sdata[blocksize];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    // Load elements to shared memory with bounds check
    int localSum = 0;
    if (i < g_size) localSum = g_idata[i];
    if (i + blockDim.x < g_size) localSum += g_idata[i + blockDim.x];
    sdata[tid] = localSum;
    __syncthreads();

    // Unroll the loop during shared memory reduction
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Perform warp-level reduction using unrolled loop
    if (tid < 32)
    {
        volatile int* smem = sdata; // Use volatile to prevent optimizing out
        smem[tid] += smem[tid + 32];
        smem[tid] += smem[tid + 16];
        smem[tid] += smem[tid + 8];
        smem[tid] += smem[tid + 4];
        smem[tid] += smem[tid + 2];
        smem[tid] += smem[tid + 1];
    }

    // Write block result to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}