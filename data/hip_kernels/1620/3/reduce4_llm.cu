#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA runtime

// Helper functions and utilities to work with CUDA

//Standard C library

#define subCOL 5248
#define COL 5248
#define ROW 358
#define WARPABLEROW 512
#define blocksize 256
#define subMatDim subCOL*WARPABLEROW
#define targetMatDim ROW * COL

__global__ void reduce4(int *g_idata, int *g_odata, int g_size)
{
    __shared__ int sdata[blocksize];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    
    // Use conditional to prevent out of bounds access
    if (i < g_size)
    {
        int sum = g_idata[i];
        if (i + blockDim.x < g_size) sum += g_idata[i + blockDim.x];
        
        sdata[tid] = sum;
    }
    else
    {
        sdata[tid] = 0;
    }
    
    __syncthreads();

    // Optimized reduction using loop unrolling
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Unroll the last warp
    if (tid < 32)
    {
        volatile int *smem = sdata; 
        smem[tid] += smem[tid + 32];
        smem[tid] += smem[tid + 16];
        smem[tid] += smem[tid + 8];
        smem[tid] += smem[tid + 4];
        smem[tid] += smem[tid + 2];
        smem[tid] += smem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}