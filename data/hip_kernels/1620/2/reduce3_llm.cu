#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA runtime

// Helper functions and utilities to work with CUDA

//Standard C library

#define subCOL 5248
#define COL 5248
#define ROW 358
#define WARPABLEROW 512
#define blocksize 256
#define subMatDim subCOL*WARPABLEROW
#define targetMatDim ROW * COL
__global__ void reduce3(int *g_idata, int *g_odata, int g_size)
{
    __shared__ int sdata[blocksize];
    
    // Load elements from global to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perfom boundary check
    if (i < g_size) {
        sdata[tid] = g_idata[i];
    } else {
        sdata[tid] = 0;
    }
    
    __syncthreads();
    
    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    
    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}