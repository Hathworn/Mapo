#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;
        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        // No probability mass or precision problems; just return the
        // first element
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithReplacement(hiprandStateMtgp32* state, int totalSamples, float* dest, long distributions, int categories, float* normDistPrefixSum) {
    for (long curDist = blockIdx.x; curDist < distributions; curDist += gridDim.x) {
        for (int sampleBase = 0; sampleBase < totalSamples; sampleBase += blockDim.y) {
            int sample = sampleBase + threadIdx.y;
            float r = 0.0f;
            
            // Load-balanced warp divergence reduction
            if (threadIdx.y == 0) {
                r = hiprand_uniform(&state[blockIdx.x]);
            }
            r = __shfl_sync(0xFFFFFFFF, r, 0); // Broadcast r to all threads in a warp
            
            if (threadIdx.x == 0 && sample < totalSamples) {
                int choice = binarySearchForMultinomial(
                    normDistPrefixSum + curDist * categories,
                    categories,
                    r
                );
                dest[curDist * totalSamples + sample] = (float)choice + 1.0f;
            }
        }
    }
}