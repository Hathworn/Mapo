#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const long size, const float maxnorm)
{
    __shared__ float buffer[32];
    long tx = threadIdx.x;
    long bx = blockIdx.x;
    long step = blockDim.x;
    float *row = data + size * bx;

    buffer[tx] = 0;

    // Calculate norm across axis
    for (long i = tx; i < size; i += step)
    {
        buffer[tx] += pow(fabs(row[i]), value);
    }

    // Reduce within block
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (tx < stride)
        {
            buffer[tx] += buffer[tx + stride];
        }
    }

    // Normalization and renormalization
    if (tx == 0)
    {
        float norm = pow(buffer[0], 1 / value);
        if (norm > maxnorm)
        {
            norm = maxnorm / (norm + 1e-7);
            buffer[0] = norm;
        }
        else
        {
            buffer[0] = 1.0f; // No renormalization needed
        }
    }

    __syncthreads();
    float normFactor = buffer[0];
    for (long i = tx; i < size; i += step)
    {
        row[i] *= normFactor;
    }
}