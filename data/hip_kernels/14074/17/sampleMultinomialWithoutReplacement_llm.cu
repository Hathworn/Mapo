#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;

        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        // No probability mass or precision problems; just return the
        // first element
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithoutReplacement(hiprandStateMtgp32* state, int totalSamples, int sample, float* dest, long distributions, int categories, float* origDist, float* normDistPrefixSum) {
    // Using shared memory to reduce global memory access
    extern __shared__ float sharedMem[];

    for (long curDistBase = blockIdx.x * blockDim.y;
         curDistBase < distributions;
         curDistBase += gridDim.x * blockDim.y) {
        long curDist = curDistBase + threadIdx.y;

        if (curDist < distributions) {
            // Load distribution into shared memory
            if (threadIdx.x < categories) {
                sharedMem[threadIdx.x] = normDistPrefixSum[curDist * categories + threadIdx.x];
            }
            __syncthreads();  // Ensure all memory loads are complete

            float r = hiprand_uniform(&state[blockIdx.x]);
            if (threadIdx.x == 0) {
                // Use shared memory for binary search
                int choice = binarySearchForMultinomial(sharedMem, categories, r);

                // Torch indices are 1-based
                dest[curDist * totalSamples + sample] = (float) choice + 1.0f;

                // Update the original probability so it is
                // not considered a second time
                origDist[curDist * categories + choice] = 0.0f;
            }
        }
    }
}