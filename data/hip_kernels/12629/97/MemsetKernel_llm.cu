#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global thread index for 1D flattened image array
    int pos = (blockIdx.y * blockDim.y + threadIdx.y) * w + (blockIdx.x * blockDim.x + threadIdx.x;

    // Only proceed if within bounds
    if (pos < w * h)
    {
        image[pos] = value; // Set image pixel value
    }
}