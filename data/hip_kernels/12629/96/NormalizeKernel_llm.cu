#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate the global thread index for both dimensions
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a conditional return to exit early for out-of-bound threads
    if (i >= h || j >= w) return;

    int pos = i * s + j;

    // Pre-compute inverse scale with a ternary operator for succinctness
    float invScale = (normalization_factor[pos] != 0.0f) ? (1.0f / normalization_factor[pos]) : 1.0f;

    // Apply normalization directly
    image[pos] *= invScale;
}