#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate global thread index
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within bounds
    if (ptidx < npoints)
    {
        const short2 loc = loc_[ptidx]; // Load data to a register

        // Perform scaling
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}