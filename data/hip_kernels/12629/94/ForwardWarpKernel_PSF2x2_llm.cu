#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate bottom left corner
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py;
    float dx = modff (cx, &px);
    float dy = modff (cy, &py);
    int tx = (int) px;
    int ty = (int) py;
    float value = src[image_row_offset + j];

    // Optimize bounds check and atomic add into a helper lambda
    auto atomicAddIfInBounds = [&](int x, int y, float weight) {
        if (!((x >= w) || (x < 0) || (y >= h) || (y < 0))) {
            _atomicAdd(dst + y * image_stride + x, value * weight);
            _atomicAdd(normalization_factor + y * image_stride + x, weight);
        }
    };

    // Fill pixels using helper
    atomicAddIfInBounds(tx, ty, dx * dy);                // Bottom right
    atomicAddIfInBounds(tx - 1, ty, (1.0f - dx) * dy);   // Bottom left
    atomicAddIfInBounds(tx - 1, ty - 1, (1.0f - dx) * (1.0f - dy)); // Upper left
    atomicAddIfInBounds(tx, ty - 1, dx * (1.0f - dy));   // Upper right
}