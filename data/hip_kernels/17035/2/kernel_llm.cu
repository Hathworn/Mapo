#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *id, float *od, int w, int h, int depth) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
    const int dataTotalSize = w * h * depth;
    const int radius = 2;
    const int filter_size = 2 * radius + 1;
    const int sW = 6;

    if (x >= w || y >= h || z >= depth) return;

    int idx = z * w * h + y * w + x;

    // Use shared memory
    __shared__ unsigned char smem[sW][sW];

    // Calculate shared memory indices with clamping
    int s_IdxY = threadIdx.y + radius;
    int s_IdxX = threadIdx.x + radius;

    // Load halo data into shared memory with boundary checks
    if (threadIdx.y < blockDim.y + 2 * radius && threadIdx.x < blockDim.x + 2 * radius) {
        int globalY = min(max(y - radius, 0), h - 1);
        int globalX = min(max(x - radius, 0), w - 1);
        int g_Idx = globalY * w + globalX;
        smem[s_IdxY][s_IdxX] = id[g_Idx];
    }
    __syncthreads();
    
    // Compute the sum using shared memory within the block
    float avg = 0.0;
    for (int i = -radius; i <= radius; i++) {
        avg += smem[s_IdxY + i][s_IdxX];
    }
    
    avg /= filter_size;

    // Write the result back to the global memory
    if (idx < dataTotalSize)
        od[idx] = avg;
}