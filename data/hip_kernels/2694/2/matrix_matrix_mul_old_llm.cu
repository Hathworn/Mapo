#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_matrix_mul_optimized(int *a, int *b, int *c, int n_row, int n_col, int n_comm)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Obtain row index using 2D grid
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Obtain col index using 2D grid

    if (row < n_row && col < n_col) { // Ensure indices are within matrix bounds
        int sum = 0;
        for (int j = 0; j < n_comm; ++j) {
            sum += a[row * n_comm + j] * b[j * n_col + col]; // Perform dot product for matrix multiplication
        }
        c[row * n_col + col] = sum; // Store result in output matrix
    }
}