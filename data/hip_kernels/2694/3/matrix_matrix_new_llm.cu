#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
Matrix Matrix multiplication with a single thread for each result element
*/

__global__ void matrix_matrix_new(int *a, int *b, int *c, int n_row, int n_col, int n_comm)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Optimize loop by calculating indices before loop
    while (tid < n_row * n_col)
    {
        int i = tid / n_col; // calculate row index once
        int j = tid % n_col; // calculate column index once

        int temp = 0;
        for (int k = 0; k < n_comm; ++k)
        {
            temp += a[i * n_comm + k] * b[j + k * n_col];
        }

        c[tid] = temp;
        
        tid += blockDim.x * gridDim.x; // Move to the next segment of work
    }
}