#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Vector addition optimized with shared memory and loop unrolling
__global__ void simple_addition(int *a, int *b, int *c, int len)
{
    // Calculate global thread ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure tid does not exceed array length
    if (tid < len) {
        // Load data into shared memory (optimization)
        __shared__ int sharedA[256]; // adjust size according to blockDim.x
        __shared__ int sharedB[256]; // adjust size according to blockDim.x
        sharedA[threadIdx.x] = a[tid];
        sharedB[threadIdx.x] = b[tid];
        __syncthreads();

        // Perform addition
        c[tid] = sharedA[threadIdx.x] + sharedB[threadIdx.x]; // optimizing memory access

        // Use loop unrolling for further performance improvement
        if (threadIdx.x + 1 < blockDim.x && tid + 1 < len)
            c[tid + 1] = a[tid + 1] + b[tid + 1];
    }
}