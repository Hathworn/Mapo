#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Optimized Vector Addition Kernel */
__global__ void good_addition(int *a, int *b, int *c, int len)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Total number of threads

    // Iterate over vector with stride
    for (int index = tid; index < len; index += stride)
    {
        c[index] = a[index] + b[index];
    }
}