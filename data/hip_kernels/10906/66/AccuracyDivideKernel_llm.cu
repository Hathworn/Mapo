#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel to handle multiple accuracy elements in parallel
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds and perform division
    if (idx < N) {
        accuracy[idx] /= N;
    }
}