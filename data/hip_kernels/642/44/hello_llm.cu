#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use threadIdx.x to display the thread ID along with the block ID
    printf("Hello world! I'm thread %d in block %d\n", threadIdx.x, blockIdx.x);
}