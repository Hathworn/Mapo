#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getPredicate_kernel(unsigned int * d_inVal, unsigned int * d_predVal, unsigned int numElems, unsigned int bitMask)
{
    unsigned int gIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (gIdx < numElems)
    {
        // Use bitwise AND and comparison to populate d_predVal
        d_predVal[gIdx] = (d_inVal[gIdx] & bitMask) ? 1 : 0;
    }
}