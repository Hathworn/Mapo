#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void incSumScanB2_kernel(unsigned int* d_outVals, unsigned int* d_inVals, size_t numVals, unsigned int* d_blockOffset) {
    unsigned int gIdx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early exit if index is out of bounds
    if (gIdx >= numVals) return;
    
    // Perform a conditional addition using ternary operator for clarity and efficiency
    unsigned int offset = (blockIdx.x > 0) ? d_blockOffset[blockIdx.x] : 0;
    d_outVals[gIdx] = d_inVals[gIdx] + offset;
}