#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reverseSort_kernel(unsigned int * d_newArray, unsigned int * d_oldArray, unsigned int numElems)
{
    unsigned int gIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (gIdx < numElems)
    {
        // Efficient calculation by using shared memory for data caching
        unsigned int reverseIdx = numElems - 1 - gIdx;
        __shared__ unsigned int s_oldArray[numElems];
        s_oldArray[gIdx] = d_oldArray[gIdx];
        __syncthreads();
        d_newArray[gIdx] = s_oldArray[reverseIdx];
    }
}