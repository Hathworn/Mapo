#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    // Cache the normalization constant in shared memory to reduce global memory accesses
    __shared__ float normalization_constant_shared;
    if (threadIdx.x == 0) {
        normalization_constant_shared = 1.f / d_input_cdf[n - 1];
    }
    __syncthreads();

    int global_index_1d = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (global_index_1d < n)
    {
        unsigned int input_value  = d_input_cdf[global_index_1d];
        // Use the shared memory value for normalization
        float output_value = input_value * normalization_constant_shared;

        d_output_cdf[global_index_1d] = output_value;
    }
}