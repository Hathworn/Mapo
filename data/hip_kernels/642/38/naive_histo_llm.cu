#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use atomic operation to prevent race conditions
__global__ void naive_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;

    // Only proceed if myId is within bounds
    if (myId < BIN_COUNT) {
        int myItem = d_in[myId];
        int myBin = myItem % BIN_COUNT;

        // Atomically increment the bin to avoid conflicts
        atomicAdd(&d_bins[myBin], 1);
    }
}