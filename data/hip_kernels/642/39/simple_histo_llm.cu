#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;
    
    // Reduce contention by using a local variable to accumulate
    __shared__ int localBins[1024];  // Assuming blockDim.x <= 1024
    if (threadIdx.x < BIN_COUNT) {
        // Initialize shared memory bins to zero
        localBins[threadIdx.x] = 0;
    }
    __syncthreads();
    
    atomicAdd(&(localBins[myBin]), 1);
    __syncthreads();
    
    if (threadIdx.x < BIN_COUNT) {
        // Aggregate localBins to global d_bins
        atomicAdd(&(d_bins[threadIdx.x]), localBins[threadIdx.x]);
    }
}