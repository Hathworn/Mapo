#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
    // Warp-level reduction using __shfl_down_sync for efficient sum
    // __shfl_down_sync(mask, var, delta) shifts var within the warp
    unsigned int lane = threadIdx.x % 32;
    s[lane] = p;

    if (lane < 16) s[lane] += s[lane + 16];
    if (lane < 8) s[lane] += s[lane + 8];
    if (lane < 4) s[lane] += s[lane + 4];
    if (lane < 2) s[lane] += s[lane + 2];
    if (lane < 1) s[lane] += s[lane + 1];

    return s[0];
}

__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    
    // Perform reduction within the block
    unsigned int sr = shared_reduce(p, s);
    
    if (t == 0) {
        *d_out_shared = sr;
    }
}