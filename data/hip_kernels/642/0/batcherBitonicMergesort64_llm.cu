#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// http://en.wikipedia.org/wiki/Bitonic_sort

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid  = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    // Iterate over sorting stages
    for (int k = 2; k <= 64; k <<= 1) 
    {
        for (int j = k >> 1; j > 0; j >>= 1) 
        {
            // Calculate the index for comparison based on thread ID
            int ixj = tid ^ j;
            // Perform comparison and conditional swap
            if (ixj > tid) 
            {
                if ((tid & k) == 0) 
                {
                    if (sdata[tid] > sdata[ixj]) 
                    {
                        float temp = sdata[tid];
                        sdata[tid] = sdata[ixj];
                        sdata[ixj] = temp;
                    }
                } 
                else 
                {
                    if (sdata[tid] < sdata[ixj]) 
                    {
                        float temp = sdata[tid];
                        sdata[tid] = sdata[ixj];
                        sdata[ixj] = temp;
                    }
                }
            }
            __syncthreads(); // Ensure all threads complete the swap before proceeding
        }
    }

    // Write result from shared memory to output
    d_out[tid] = sdata[tid];
}