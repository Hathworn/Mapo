#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void swapVals_kernel(unsigned int * d_newArray, unsigned int * d_oldArray, unsigned int numElems)
{
    unsigned int gIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Using shared memory to reduce global memory access
    __shared__ unsigned int shared_oldArray[256]; // Assuming blockDim.x <= 256

    if (gIdx < numElems)
    {
        // Load data into shared memory
        shared_oldArray[threadIdx.x] = d_oldArray[gIdx];
        __syncthreads();

        // Store data from shared memory to global memory
        d_newArray[gIdx] = shared_oldArray[threadIdx.x];
    }
}