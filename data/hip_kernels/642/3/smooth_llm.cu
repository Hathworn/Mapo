#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth(float * v_new, const float * v) {
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;
    
    // Ensure indices are within bounds
    if (myIdx < numThreads) {
        int myLeftIdx = (myIdx == 0) ? 0 : myIdx - 1;
        int myRightIdx = (myIdx == (numThreads - 1)) ? numThreads - 1 : myIdx + 1;
        
        // Use shared memory to reduce global memory accesses
        extern __shared__ float shared[];
        shared[threadIdx.x] = v[myIdx];
        
        if (threadIdx.x > 0 && myIdx > 0) {
            shared[threadIdx.x - 1] = v[myIdx - 1];
        }
        if (threadIdx.x < blockDim.x - 1 && myIdx < numThreads - 1) {
            shared[threadIdx.x + 1] = v[myIdx + 1];
        }
        __syncthreads();
        
        float myElt = shared[threadIdx.x];
        float myLeftElt = (threadIdx.x > 0) ? shared[threadIdx.x - 1] : v[myLeftIdx];
        float myRightElt = (threadIdx.x < blockDim.x - 1) ? shared[threadIdx.x + 1] : v[myRightIdx];
        
        // Compute the smooth value
        v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
    }
}