#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void incSumScanB1_kernel(unsigned int* d_outVals, unsigned int* d_inVals, size_t numVals, unsigned int* d_blockOffset, unsigned int valOffset)
{
    unsigned int tIdx = threadIdx.x;
    unsigned int gIdx = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ unsigned int s_incScan[];

    if (gIdx >= numVals) return;
    
    // Load input plus offset for first thread in block
    s_incScan[tIdx] = d_inVals[gIdx] + ((tIdx == 0) ? valOffset : 0);
    __syncthreads();

    // Inclusive scan within block, using shared memory
    for (int offset = 1; offset < blockDim.x; offset *= 2)
    {
        unsigned int neighbor = (tIdx >= offset) ? s_incScan[tIdx - offset] : 0;
        __syncthreads(); // Ensure all threads have retrieved their neighbors
        s_incScan[tIdx] += neighbor;
        __syncthreads(); // Ensure all additions complete before next loop
    }

    d_outVals[gIdx] = s_incScan[tIdx];

    // Store block's cumulative sum (last thread's result) in blockOffset array
    if (tIdx == blockDim.x - 1 && blockIdx.x + 1 < gridDim.x)
    {
        d_blockOffset[blockIdx.x + 1] = s_incScan[tIdx];
    }
}