#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void hello()
{
    int idx = threadIdx.x; // Cache thread index to a local variable for potential compiler optimization
    printf("Hello world! I'm thread %d\n", idx);
}