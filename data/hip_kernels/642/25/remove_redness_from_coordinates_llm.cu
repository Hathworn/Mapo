#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void remove_redness_from_coordinates( const unsigned int*  d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, int    num_coordinates, int    num_pixels_y, int    num_pixels_x, int    template_half_height, int    template_half_width )
{
    int nx = num_pixels_x;
    int ny = num_pixels_y;
    int global_index_1d = ( blockIdx.x * blockDim.x ) + threadIdx.x;

    int imgSize = nx * ny;

    if ( global_index_1d < num_coordinates )
    {
        unsigned int image_index_1d = d_coordinates[ imgSize - global_index_1d - 1 ];
        ushort2 image_index_2d = make_ushort2(image_index_1d % nx, image_index_1d / nx);

        int y_start = max(0, image_index_2d.y - template_half_height); // Clamp start y
        int y_end = min(ny - 1, image_index_2d.y + template_half_height); // Clamp end y
        int x_start = max(0, image_index_2d.x - template_half_width); // Clamp start x
        int x_end = min(nx - 1, image_index_2d.x + template_half_width); // Clamp end x

        for ( int y = y_start; y <= y_end; y++ )
        {
            for ( int x = x_start; x <= x_end; x++ )
            {
                int image_offset_index_1d_clamped = ( nx * y ) + x; // Precomputed index

                unsigned char g_value = d_g[ image_offset_index_1d_clamped ];
                unsigned char b_value = d_b[ image_offset_index_1d_clamped ];

                unsigned int gb_average = ( g_value + b_value ) / 2;

                d_r_output[ image_offset_index_1d_clamped ] = (unsigned char)gb_average;
            }
        }
    }
}