#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void incSumScan_kernel(unsigned int* d_outVals, unsigned int* d_inVals, size_t numVals)
{
    unsigned int tIdx = threadIdx.x;
    unsigned int gIdx = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ unsigned int s_incScan[];

    // Return early if global index exceeds number of values
    if (gIdx >= numVals) return;

    s_incScan[tIdx] = (gIdx < numVals) ? d_inVals[gIdx] : 0;
    __syncthreads();

    // Use more efficient loop for shared memory scan
    for (unsigned int offset = 1; offset < blockDim.x; offset *= 2)
    {
        unsigned int temp = 0;
        if (tIdx >= offset)
            temp = s_incScan[tIdx - offset];

        __syncthreads();
        s_incScan[tIdx] += temp;
        __syncthreads();
    }

    // Store result to global output array if within bounds
    if (gIdx < numVals) 
        d_outVals[gIdx] = s_incScan[tIdx];
}