#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lumHistExclusiveScan_kernel(unsigned int *d_out, unsigned int *d_in, int numItems)
{
    extern __shared__ unsigned int s_exScan[];
    int tid = threadIdx.x;

    // Load input into shared memory, use a single conditional statement
    s_exScan[tid] = (tid > 0) ? d_in[tid - 1] : 0;
    __syncthreads();

    // Use a reduced number of synchronizations by restructuring the loop
    for (int offset = 1; offset < numItems; offset *= 2) {
        unsigned int temp = s_exScan[tid];
        if ((tid - offset) >= 0) {
            s_exScan[tid] = temp + s_exScan[tid - offset];
        }
        __syncthreads();
    }
    // Write the result back to global memory
    d_out[tid] = s_exScan[tid];
}