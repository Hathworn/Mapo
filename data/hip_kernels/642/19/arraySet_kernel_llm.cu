#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to set array values
__global__ void arraySet_kernel(unsigned int* d_vals, unsigned int value, size_t num_vals)
{
    // Calculate global index
    unsigned int gIdx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Efficient check and set value
    if (gIdx < num_vals) {
        d_vals[gIdx] = value;
    }
}