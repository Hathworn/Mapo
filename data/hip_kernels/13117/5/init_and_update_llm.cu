#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_and_update(float *values_d, int tpoints, int nsteps) {
    int idx = threadIdx.x + blockIdx.x * BLOCK_SIZE;

    if (idx <= 1 || idx >= tpoints)
        return;

    float old_v, v, new_v;
    float x, tmp;
    tmp = tpoints - 1;
    x = (float)(idx - 1) / tmp;

    v = sinf(2.0f * PI * x);  // Use sinf for improved performance with float
    old_v = v;

    float coeff = 0.82f;  // Precompute constant component for efficiency

    for (int i = 1; i <= nsteps; i++) {
        new_v = coeff * v - old_v;
        old_v = v;
        v = new_v;
    }

    values_d[idx] = v;
}