#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_and_update (float *values_d, int tpoints, int nsteps) {
    int idx = threadIdx.x + blockIdx.x * BLOCK_SIZE;

    if (idx <= 1 || idx >= tpoints)
        return;

    float old_v, v, new_v;
    float x = (float)(idx - 1) / (tpoints - 1); // Precompute temporary value as x

    v = sinf(2.0f * PI * x); // Use sinf for single precision performance
    old_v = v;

    float neg_coef_v = -0.18f * v; // Precompute constant multiplication outside the loop

    for (int i = 1; i <= nsteps; i++) {
        new_v = 2.0f * v - old_v + neg_coef_v; // Simplified arithmetic
        old_v = v;
        v = new_v;
    }

    values_d[idx] = v;
}