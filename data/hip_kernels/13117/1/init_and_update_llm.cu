#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAXPOINTS 1000000
#define MAXSTEPS 1000000
#define MINPOINTS 20
#define PI 3.14159265

#define BLOCK_SIZE 512

void check_param(void);
void printfinal (void);

/**********************************************************************
*     Initialize points on line
*********************************************************************/

/**********************************************************************
*     Update all values along line a specified number of times
*********************************************************************/
__global__ void init_and_update (float *values_d, int tpoints, int nsteps){
    int idx = threadIdx.x + blockIdx.x * BLOCK_SIZE;
    
    // Ensure index is within the valid range
    if (idx < 1 || idx >= tpoints - 1)
        return;

    float old_v, v, new_v;

    // Calculate position and initial value using sin function
    float x = static_cast<float>(idx) / (tpoints - 1);
    v = sinf(2.0f * PI * x);
    old_v = v;

    // Calculate wave propagation through iterations
    for (int i = 0; i < nsteps; i++) {
        new_v = 2.0f * v - old_v + 0.09f * (-2.0f * v);
        old_v = v;
        v = new_v;
    }

    // Store the final computed value
    values_d[idx] = v;
}