#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
int row = 0;
int col = 0;
using namespace std;

__global__ void gpu_transpose(float *dst, float *A, int col, int row) {
    // Calculate the global index for the row and column
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Use threads to handle both dimensions for better parallelism
    if(x < col && y < row) {
        dst[y * col + x] = A[x * row + y];
    }
}
```
