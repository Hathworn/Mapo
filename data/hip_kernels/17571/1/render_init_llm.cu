#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

__global__ void render_init(int max_x, int max_y, hiprandState* rand_state) {
    // Calculate the global index
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    // Check index bounds
    if ((i < max_x) && (j < max_y)) {
        // Simplify calculations for unique seed index
        int k = (i + j) % 100;

        if ((i + j) / 100 == 0) {
            // Initialize random state with unique k
            hiprand_init(1995, k, 0, &rand_state[k]);
        }
    }
}