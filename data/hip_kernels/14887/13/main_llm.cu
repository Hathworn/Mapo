#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "initGuessBuffers_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    const uchar4 *srcImg = NULL;
hipMalloc(&srcImg, XSIZE*YSIZE);
float3 *guess1 = NULL;
hipMalloc(&guess1, XSIZE*YSIZE);
float3 *guess2 = NULL;
hipMalloc(&guess2, XSIZE*YSIZE);
const uint nRows = 1;
const uint nCols = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
initGuessBuffers<<<gridBlock, threadBlock>>>(srcImg,guess1,guess2,nRows,nCols);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
initGuessBuffers<<<gridBlock, threadBlock>>>(srcImg,guess1,guess2,nRows,nCols);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
initGuessBuffers<<<gridBlock, threadBlock>>>(srcImg,guess1,guess2,nRows,nCols);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}