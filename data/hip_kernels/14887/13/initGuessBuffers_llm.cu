#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initGuessBuffers(const uchar4* srcImg, float3* guess1, float3* guess2, const uint nRows, const uint nCols) {
    const uint nSamps = nRows * nCols;
    
    const uint samp = threadIdx.x + blockDim.x * blockIdx.x;
    if (samp < nSamps) {
        uchar4 pixel = srcImg[samp]; // Load once to avoid redundant global memory reads

        guess1[samp] = make_float3(pixel.x, pixel.y, pixel.z); // Reduced code duplication
        guess2[samp] = make_float3(pixel.x, pixel.y, pixel.z);
    }
}