#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use blockIdx.x and blockDim.x for better identification of threads in grid
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello world! I'm thread %d\n", globalThreadId);
}