#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
    // Load the value into shared memory
    s[threadIdx.x] = p;
    __syncthreads();

    // Perform reduction in shared memory
    if (threadIdx.x < 16) s[threadIdx.x] += s[threadIdx.x + 16];
    __syncthreads();
    if (threadIdx.x < 8) s[threadIdx.x] += s[threadIdx.x + 8];
    __syncthreads();
    if (threadIdx.x < 4) s[threadIdx.x] += s[threadIdx.x + 4];
    __syncthreads();
    if (threadIdx.x < 2) s[threadIdx.x] += s[threadIdx.x + 2];
    __syncthreads();
    if (threadIdx.x < 1) s[threadIdx.x] += s[threadIdx.x + 1];
    __syncthreads();

    return s[0]; // Return result from s[0]
}

__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in)
{
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    if (t == 0)
    {
        *d_out_shared = sr;
    }
}