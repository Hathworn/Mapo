#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tonemap(float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new,
                        float min_log_Y, float max_log_Y, float log_Y_range, int num_bins, int num_pixels_y, int num_pixels_x) {
    int nx = num_pixels_x;
    int ny = num_pixels_y;

    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < nx && y < ny) {
        int image_index_1d = (nx * y) + x;

        float x_val = d_x[image_index_1d];
        float y_val = d_y[image_index_1d];
        float log_Y = d_log_Y[image_index_1d];

        int bin_index = min(num_bins - 1, int((num_bins * (log_Y - min_log_Y)) / log_Y_range));
        float Y_new = d_cdf_norm[bin_index];

        float Y_over_y = Y_new / y_val; // Precompute repeated division
        float X_new = x_val * Y_over_y;
        float Z_new = (1 - x_val - y_val) * Y_over_y;

        // Consolidate color transformation calculations
        float r_new = (X_new * 3.2406f) + (Y_new * -1.5372f) + (Z_new * -0.4986f);
        float g_new = (X_new * -0.9689f) + (Y_new * 1.8758f) + (Z_new * 0.0415f);
        float b_new = (X_new * 0.0557f) + (Y_new * -0.2040f) + (Z_new * 1.0570f);

        d_r_new[image_index_1d] = r_new;
        d_g_new[image_index_1d] = g_new;
        d_b_new[image_index_1d] = b_new;
    }
}