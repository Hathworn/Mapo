#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void remove_redness_from_coordinates(const unsigned int* d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, int num_coordinates, int num_pixels_y, int num_pixels_x, int template_half_height, int template_half_width)
{
    int ny = num_pixels_y;
    int nx = num_pixels_x;
    int global_index_1d = (blockIdx.x * blockDim.x) + threadIdx.x;

    int imgSize = num_pixels_x * num_pixels_y;

    if (global_index_1d < num_coordinates)
    {
        unsigned int image_index_1d = d_coordinates[imgSize - global_index_1d - 1];
        ushort2 image_index_2d = make_ushort2(image_index_1d % num_pixels_x, image_index_1d / num_pixels_x);
        
        // Precalculate boundaries to optimize loop range checks
        int start_y = max(0, image_index_2d.y - template_half_height);
        int end_y = min(ny - 1, image_index_2d.y + template_half_height);
        int start_x = max(0, image_index_2d.x - template_half_width);
        int end_x = min(nx - 1, image_index_2d.x + template_half_width);
        
        for (int y = start_y; y <= end_y; y++)
        {
            int base_y_index = nx * y;  // Calculate base index for y coordinate once per loop
            for (int x = start_x; x <= end_x; x++)
            {
                int image_offset_index_1d_clamped = base_y_index + x;

                unsigned char g_value = d_g[image_offset_index_1d_clamped];
                unsigned char b_value = d_b[image_offset_index_1d_clamped];

                unsigned int gb_average = (g_value + b_value) / 2;

                d_r_output[image_offset_index_1d_clamped] = (unsigned char)gb_average;
            }
        }
    }
}