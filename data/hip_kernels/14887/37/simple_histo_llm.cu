#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // Ensure the thread handles only valid data index
    if (tid < BIN_COUNT)
    {
        int item = d_in[tid];
        int bin = item % BIN_COUNT;

        // Coalesce atomic operations by using a shared memory buffer
        extern __shared__ int shared_bins[];
        atomicAdd(&shared_bins[bin], 1);

        // Synchronize threads to ensure all updates are completed
        __syncthreads();
        
        // Gather results from shared memory to global memory
        atomicAdd(&d_bins[bin], shared_bins[bin]);
    }
}