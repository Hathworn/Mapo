#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void batcherBitonicMergesort64(float *d_out, const float *d_in)
{
    // Shared memory allocation
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    // Bitonic sort algorithm stages
    for (int stage = 0; stage <= 5; stage++)
    {
        int k = 1 << stage; // Calculate power of two

        for (int substage = stage; substage >= 0; substage--)
        {
            int j = 1 << substage; // Calculate current substage offset
            int xorIdx = tid ^ j;  // Calculate paired index using XOR

            if (xorIdx > tid)
            {
                if ((tid & k) == 0)
                {
                    // Compare and swap in ascending order
                    if (sdata[tid] > sdata[xorIdx])
                    {
                        float temp = sdata[tid];
                        sdata[tid] = sdata[xorIdx];
                        sdata[xorIdx] = temp;
                    }
                }
                else
                {
                    // Compare and swap in descending order
                    if (sdata[tid] < sdata[xorIdx])
                    {
                        float temp = sdata[tid];
                        sdata[tid] = sdata[xorIdx];
                        sdata[xorIdx] = temp;
                    }
                }
            }
            __syncthreads(); // Ensure all threads complete stage/substage
        }
    }

    // Write results to output
    d_out[tid] = sdata[tid];
}