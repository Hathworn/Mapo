#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
    extern __shared__ float sdata[]; // shared memory for in-block reduction
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // Load input into shared memory
    sdata[tid] = d_in[myId];
    __syncthreads(); // ensure all loads are done

    // Optimize reduction by keeping data in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // ensure all adds are done
    }

    // Write result for this block to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}