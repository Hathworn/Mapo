#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel by using warp-synchronous printf
__global__ void hello()
{
    // Calculate the unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single thread to print the message for the block
    if (threadIdx.x == 0) {
        printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
    }
}