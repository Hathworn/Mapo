#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth(float *v_new, const float *v) {
    // Calculate global thread index using built-in variables
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
    // Optimize boundary index handling
    int numThreads = blockDim.x * gridDim.x;
    float myElt = v[myIdx];
    float myLeftElt = (myIdx > 0) ? v[myIdx - 1] : v[0];
    float myRightElt = (myIdx < numThreads - 1) ? v[myIdx + 1] : v[numThreads - 1];
    // Efficiently compute smoothed element
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
}