#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void odd(int *darr, int n) {
    int k = threadIdx.x * 2 + 1; // Calculate index once
    if (k <= n - 2) {
        int current = darr[k];
        int next = darr[k + 1];
        if (current > next) {
            darr[k] = next;
            darr[k + 1] = current;
        }
    }
}