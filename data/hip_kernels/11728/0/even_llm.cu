#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void even(int *darr, int n) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    int t;
    k = k * 2;
    if (k <= n - 2) {
        // Use one conditional statement to check and swap elements
        if (darr[k] > darr[k + 1]) {
            t = darr[k];
            darr[k] = darr[k + 1];
            darr[k + 1] = t;
        }
    }
}
```
