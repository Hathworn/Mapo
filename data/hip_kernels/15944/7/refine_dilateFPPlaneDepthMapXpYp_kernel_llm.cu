#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_dilateFPPlaneDepthMapXpYp_kernel(float* fpPlaneDepthMap, int fpPlaneDepthMap_p, float* maskMap, int maskMap_p, int width, int height, int xp, int yp, float fpPlaneDepth)
{
    // Use shared memory to reduce global memory accesses
    __shared__ float sharedMask[1024]; // Adjust size as needed

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int threadId = threadIdx.y * blockDim.x + threadIdx.x;

    // Check bounds and load data into shared memory
    if (x < width && y < height) {
        sharedMask[threadId] = maskMap[y * maskMap_p + x];
        __syncthreads(); // Ensure all loads are done before proceeding

        if ((x + xp >= 0) && (y + yp >= 0) && (x + xp < width) && (y + yp < height)) {
            // Use value from shared memory
            if (sharedMask[threadId] > 0.0f) {
                fpPlaneDepthMap[(y + yp) * fpPlaneDepthMap_p + (x + xp)] = fpPlaneDepth;
            }
        }
    }
}