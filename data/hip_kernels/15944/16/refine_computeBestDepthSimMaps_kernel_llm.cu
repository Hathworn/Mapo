#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float refineDepthSubPixel(const float3& depths, const float3& sims)
{
    float outDepth = -1.0f;

    float simM1 = (sims.x + 1.0f) * 0.5f;
    float simP1 = (sims.z + 1.0f) * 0.5f;
    float sim1 = (sims.y + 1.0f) * 0.5f;

    if ((simM1 > sim1) && (simP1 > sim1))
    {
        // Calculate displacement step using quadratic interpolation
        float dispStep = -((simP1 - simM1) / (2.0f * (simP1 + simM1 - 2.0f * sim1)));

        float floatDepthM1 = depths.x;
        float floatDepthP1 = depths.z;

        // Calculate refined depth using linear function fit
        float b = (floatDepthP1 + floatDepthM1) * 0.5f;
        float a = b - floatDepthM1;
        outDepth = a * dispStep + b;
    }

    return outDepth;
}

__global__ void refine_computeBestDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float3* isims, int isims_p, float3* idpts, int idpts_p, int width, int height, float simThr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < width) && (y < height))
    {
        float3 depths = idpts[y * idpts_p + x];
        float3 sims = isims[y * isims_p + x];

        // Use ternary operator for initial comparison of depths and sims
        float outDepth = (sims.x < sims.y) ? depths.x : depths.y;
        float outSim = (sims.x < sims.y) ? sims.x : sims.y;
        if (sims.z < outSim) {
            outDepth = depths.z;
            outSim = sims.z;
        }

        // Refine depth further with subpixel refinement
        float refinedDepth = refineDepthSubPixel(depths, sims);
        if (refinedDepth > 0.0f)
        {
            outDepth = refinedDepth;
        }

        // Set output based on similarity threshold
        osim[y * osim_p + x] = (outSim < simThr) ? outSim : 1.0f;
        odpt[y * odpt_p + x] = (outSim < simThr) ? outDepth : -1.0f;
    }
}