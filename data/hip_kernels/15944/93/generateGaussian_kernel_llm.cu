#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generateGaussian_kernel(float* og, float delta, int radius)
{
    // Calculate once and reuse for efficiency
    float varFactor = 1.0f / (2 * delta * delta);
    
    // Use blockIdx.x to compute global index for flexibility with larger arrays
    int x = threadIdx.x - radius;
    og[threadIdx.x] = __expf(-(x * x) * varFactor);
}