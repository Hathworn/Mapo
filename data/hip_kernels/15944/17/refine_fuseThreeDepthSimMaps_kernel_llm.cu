#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_fuseThreeDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float* isimLst, int isimLst_p, float* idptLst, int idptLst_p, float* isimAct, int isimAct_p, float* idptAct, int idptAct_p, int width, int height, float simThr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < width) && (y < height)) {
        float dpts[3];
        float sims[3];

        // Load memory directly to registers
        dpts[0] = odpt[y * odpt_p + x];
        dpts[1] = idptLst[y * idptLst_p + x];
        dpts[2] = idptAct[y * idptAct_p + x];

        sims[0] = osim[y * osim_p + x];
        sims[1] = isimLst[y * isimLst_p + x];
        sims[2] = isimAct[y * isimAct_p + x];

        // Simplify calculations using a single pass
        float outDpt = dpts[0];
        float outSim = sims[0];

        if (sims[1] < outSim) {
            outDpt = dpts[1];
            outSim = sims[1];
        }
        
        if (sims[2] < outSim) {
            outDpt = dpts[2];
            outSim = sims[2];
        }

        osim[y * osim_p + x] = outSim;
        odpt[y * odpt_p + x] = outDpt;
    }
}