#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

__global__ void gameOfLifeKernel(unsigned char* d_src, unsigned char* d_dst, const size_t width, const size_t height) {
    extern __shared__ unsigned char board_sh[];

    size_t glob_x = blockDim.x * blockIdx.x + threadIdx.x;
    size_t glob_y = blockDim.y * blockIdx.y + threadIdx.y;
    size_t glob_idx = glob_y * width + glob_x;

    size_t index = blockDim.x * threadIdx.y + threadIdx.x;

    int share_width = blockDim.x + 2;
    int share_height = blockDim.y + 2;
    
    bool isActive = (glob_x < width && glob_y < height);

    // Optimized shared memory copying using contiguous memory access
    for (int share_idx = index; share_idx < share_width * share_height; share_idx += (blockDim.x * blockDim.y)) {
        int x_img = (blockDim.x * blockIdx.x - 1) + (share_idx % share_width);
        int y_img = (blockDim.y * blockIdx.y - 1) + (share_idx / share_width);

        x_img = (x_img + width) % width; // Wrap around logic
        y_img = (y_img + height) % height; // Wrap around logic

        board_sh[share_idx] = d_src[width * y_img + x_img];
    }

    __syncthreads();

    if (isActive) {
        unsigned char me = board_sh[share_width * (threadIdx.y + 1) + threadIdx.x + 1];

        // Count neighbors from shared memory
        int count = 0;
        count += board_sh[share_width * (threadIdx.y) + threadIdx.x];
        count += board_sh[share_width * (threadIdx.y) + threadIdx.x + 1];
        count += board_sh[share_width * (threadIdx.y) + threadIdx.x + 2];
        count += board_sh[share_width * (threadIdx.y + 1) + threadIdx.x];
        count += board_sh[share_width * (threadIdx.y + 1) + threadIdx.x + 2];
        count += board_sh[share_width * (threadIdx.y + 2) + threadIdx.x];
        count += board_sh[share_width * (threadIdx.y + 2) + threadIdx.x + 1];
        count += board_sh[share_width * (threadIdx.y + 2) + threadIdx.x + 2];

        // Game of life rules applied
        if (me == 1) {
            d_dst[glob_idx] = (count == 2 || count == 3) ? 1 : 0;
        } else {
            d_dst[glob_idx] = (count == 3) ? 1 : 0;
        }
    }
}