#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use consistent indexing
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i < max_x && j < max_y) { // Use boundary condition check early
        int pixel_index = j * max_x + i;
        // Each thread gets same seed, a different sequence number, no offset
        hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
    }
}