#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rgb2gray(float *grayImage, float *rgbImage, int channels, int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure thread is within image bounds
    if (x < width && y < height) {
        // Compute 1D coordinate for the grayscale image
        int grayOffset = y * width + x;
        int rgbOffset = grayOffset * channels;

        // Cache RGB values to registers for faster access
        float r = rgbImage[rgbOffset];     
        float g = rgbImage[rgbOffset + 1];
        float b = rgbImage[rgbOffset + 2];

        // Convert and store grayscale value
        grayImage[grayOffset] = 0.21f * r + 0.71f * g + 0.07f * b;
    }
}