#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float * __restrict__ A, const float * __restrict__ B, float * __restrict__ C, int numElements)
{
    // Calculate the global index for each thread
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Use a single conditional branch to avoid divergent execution
    if (i < numElements)
    {
        // Compute the vector addition
        C[i] = A[i] + B[i];
    }
}