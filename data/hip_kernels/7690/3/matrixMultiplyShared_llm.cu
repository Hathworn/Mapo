#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16

__global__ void matrixMultiplyShared(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
    // Declare shared memory for tiles of A and B
    __shared__ float sharedA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float sharedB[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    float value = 0.0;

    // Loop over tiles of matrices A and B
    for (int tileIdx = 0; tileIdx < (numAColumns + BLOCK_SIZE - 1) / BLOCK_SIZE; ++tileIdx) {
        // Load data into shared memory if within bounds
        if (row < numARows && tileIdx * BLOCK_SIZE + threadIdx.x < numAColumns)
            sharedA[threadIdx.y][threadIdx.x] = A[row * numAColumns + tileIdx * BLOCK_SIZE + threadIdx.x];
        else
            sharedA[threadIdx.y][threadIdx.x] = 0.0;
        
        if (col < numBColumns && tileIdx * BLOCK_SIZE + threadIdx.y < numBRows)
            sharedB[threadIdx.y][threadIdx.x] = B[(tileIdx * BLOCK_SIZE + threadIdx.y) * numBColumns + col];
        else
            sharedB[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();  // Synchronize threads within a block

        // Compute local dot product
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            value += sharedA[threadIdx.y][k] * sharedB[k][threadIdx.x];
        }

        __syncthreads();  // Synchronize threads before loading new tiles
    }

    // Write value to C if within bounds
    if (row < numCRows && col < numCColumns) {
        C[row * numCColumns + col] = value;
    }
}