#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blurKernel(float *out, const float *in, int width, int height) {
    // Calculate global thread coordinates
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        float pixVal = 0.0f;
        int pixels = 0;

        // Optimize loop by using registers for constants
        int blurSize = BLUR_SIZE;
        
        // Unroll loops for better performance
        #pragma unroll
        for (int blurrow = -blurSize; blurrow <= blurSize; ++blurrow) {
            #pragma unroll
            for (int blurcol = -blurSize; blurcol <= blurSize; ++blurcol) {
                int currow = row + blurrow;
                int curcol = col + blurcol;
                
                // Verify valid image pixel using boundary checks
                if (currow >= 0 && currow < height && curcol >= 0 && curcol < width) {
                    pixVal += in[currow * width + curcol];
                    pixels++;
                }
            }
        }

        // Write new pixel value
        out[row * width + col] = pixVal / pixels;
    }
}