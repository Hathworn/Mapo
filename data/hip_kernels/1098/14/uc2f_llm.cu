#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifndef M_PI
#define M_PI 3.14159265358979323846f
#endif

#define MAX_SPOTS 1024
#define BLOCK_SIZE 256
#define SLM_SIZE 512
#if ((SLM_SIZE==16)||(SLM_SIZE==32)||(SLM_SIZE==64)||(SLM_SIZE==128)||(SLM_SIZE==256)||(SLM_SIZE==512)||(SLM_SIZE==1024)||(SLM_SIZE==2048))
#define SLMPOW2
#endif

__global__ void uc2f(float *f, unsigned char *uc, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure only needed threads run the computation, this can help prevent wasted resources
    if (idx < N) {
        // Precompute constant to avoid redundant calculation and reduce instruction count
        const float coeff = 2.0f * M_PI / 256.0f;
        f[idx] = uc[idx] * coeff - M_PI;
    }
}