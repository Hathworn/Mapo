#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifndef M_PI
#define M_PI 3.14159265358979323846f
#endif

#define MAX_SPOTS 1024
#define BLOCK_SIZE 256
#define SLM_SIZE 512
#if ((SLM_SIZE==16)||(SLM_SIZE==32)||(SLM_SIZE==64)||(SLM_SIZE==128)||(SLM_SIZE==256)||(SLM_SIZE==512)||(SLM_SIZE==1024)||(SLM_SIZE==2048))
#define SLMPOW2
#endif

__device__ unsigned char applyPolLUT(float phase2pi, float X, float Y, float *s_c) {
    // Calculate polynomial lookup table value based on input variables and coefficients
    switch (c_N_PolLUTCoeff[0]) {
    case 120:
        return (unsigned char)(s_c[0] + s_c[1]*X + /* values omitted for brevity */ + s_c[119]*phase2pi*phase2pi*phase2pi);
    case 84:
        return (unsigned char)(s_c[0] + s_c[1]*X + /* values omitted for brevity */ + s_c[83]*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi);
    case 56:
        return (unsigned char)(s_c[0] + s_c[1]*X + /* values omitted for brevity */ + s_c[55]*phase2pi*phase2pi*phase2pi*phase2pi*phase2pi);
    case 35:
        return (unsigned char)(s_c[0] + s_c[1]*X + /* values omitted for brevity */ + s_c[34]*phase2pi*phase2pi*phase2pi*phase2pi);
    case 20:
        return (unsigned char)(s_c[0] + s_c[1]*X + /* values omitted for brevity */ + s_c[19]*phase2pi*phase2pi*phase2pi);
    default:
        return 0;
    }
}

__device__ int getYint(int index, int X_int) {
    // Calculate the Y integer index
#ifdef SLMPOW2
    return (index-X_int)>>c_log2data_w[0];
#else
    return (int)(floor((float)index/c_data_w_f[0]));
#endif
}

__device__ int getXint(int index) {
    // Calculate the X integer index
#ifdef SLMPOW2
    return index & (c_data_w[0]-1);
#else
    return index % c_data_w[0];
#endif
}

__device__ float ApplyAberrationCorrection(float pSpot, float correction) {
    // Correct the spot phase
    pSpot = pSpot - correction;
    return (pSpot - (2.0f * M_PI) * floor((pSpot + M_PI) / (2.0f * M_PI)));
}

__device__ int phase2int32(float phase2pi) {
    // Convert phase to integer
    return (int)floor((phase2pi + M_PI) * 256.0f / (2.0f * M_PI));
}

__device__ unsigned char phase2uc(float phase2pi) {
    // Convert phase to unsigned char
    return (unsigned char)floor((phase2pi + M_PI) * 256.0f / (2.0f * M_PI));
}

__device__ float uc2phase(float uc) {
    // Convert unsigned char to phase
    return (float)uc * 2.0f * M_PI / 256.0f - M_PI;
}

__global__ void ApplyCorrections(unsigned char *g_pSLM_uc, unsigned char *g_LUT, float *g_AberrationCorr_f, float *g_LUTPolCoeff_f) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    float pSLM2pi_f = uc2phase(g_pSLM_uc[idx]);
    
    // Apply aberration correction if needed
    if (c_useAberrationCorr_b[0])
        pSLM2pi_f = ApplyAberrationCorrection(pSLM2pi_f, g_AberrationCorr_f[idx]);
    
    if (c_usePolLUT_b[0]) {
        // Load polynomial coefficients into shared memory
        int X_int = getXint(idx);
        int Y_int = getYint(idx, X_int);
        float X = c_SLMpitch_f[0] * (X_int - c_half_w_f[0]);
        float Y = c_SLMpitch_f[0] * (Y_int - c_half_w_f[0]);
        __shared__ float s_LUTcoeff[120];
        
        if (tid < c_N_PolLUTCoeff[0])
            s_LUTcoeff[tid] = g_LUTPolCoeff_f[tid];
        __syncthreads();
        
        // Apply polynomial LUT
        g_pSLM_uc[idx] = applyPolLUT(pSLM2pi_f, X, Y, s_LUTcoeff);
    } else if (c_applyLUT_b[0]) {
        // Load LUT into shared memory
        __shared__ unsigned char s_LUT[256];
        
        if (tid < 256)
            s_LUT[tid] = g_LUT[tid];
        __syncthreads();
        
        // Apply LUT
        g_pSLM_uc[idx] = s_LUT[phase2int32(pSLM2pi_f)];
    } else {
        // Convert phase to unsigned char without LUT
        g_pSLM_uc[idx] = phase2uc(pSLM2pi_f);
    }
}