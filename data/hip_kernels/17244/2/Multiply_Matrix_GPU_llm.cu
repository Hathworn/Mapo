#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Multiply_Matrix_GPU(float* A, float* B, float* C, int BLOCK_SIZE, int N) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float total = 0.0;
    int ia = N * BLOCK_SIZE * by + ty;  // Offset for matrix A
    int ib = BLOCK_SIZE * bx + tx;      // Offset for matrix B

    // Use a wider stride for the loop to ensure coalesced access
    for (int k = 0; k < N; k += BLOCK_SIZE) {
        // Load a block strip of A and B into shared memory
        __shared__ float As[BLOCK_SIZE];
        __shared__ float Bs[BLOCK_SIZE];

        // Each thread loads one element into shared memory
        As[ty] = A[ia + k + ty * N];
        Bs[tx] = B[ib + k * N];

        __syncthreads(); // Ensure all data is loaded into shared memory

        // Multiply and accumulate
        for (int n = 0; n < BLOCK_SIZE; ++n) {
            total += As[n] * Bs[n];
        }

        __syncthreads(); // Ensure all calculations are done before next load
    }

    int ic = N * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    // Store the result in matrix C
    C[ic + N * ty + tx] = total;
}