#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Pi_GPU(float *x, float *y, int *totalCounts, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // calculate element index
    int threadCount = gridDim.x * blockDim.x; // calculate step size

    int countPoints = 0;
    // Optimize loop by moving frequently accessed values to registers
    float xVal, yVal;
    for (int i = idx; i < N; i += threadCount) {
        xVal = x[i];
        yVal = y[i];
        if (xVal * xVal + yVal * yVal < 1.0f) {
            countPoints++;
        }
    }
    atomicAdd(totalCounts, countPoints); // accumulate each thread's result
}