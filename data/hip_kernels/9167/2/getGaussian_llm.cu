#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void generate2DGaussian(double *output, double sigma, int sz, bool normalize) {
    // Compute coordinates and index
    const int colIdx = threadIdx.x;
    const int rowIdx = threadIdx.y;
    int linearIdx = rowIdx * sz + colIdx;

    // Calculate distance from center and assign Gaussian value
    int distx = colIdx - sz / 2;
    int disty = rowIdx - sz / 2;
    double gaussValue = exp(-(distx * distx + disty * disty) / (2.0 * sigma * sigma));
    output[linearIdx] = gaussValue;

    // Optional normalization
    if (normalize) {
        // Synchronize threads before summing
        __syncthreads();

        // Optimize sum calculation using a single thread
        if (linearIdx == 0) {
            double sum = 0.0;
            for (int i = 0; i < sz * sz; ++i) {
                sum += output[i];
            }
            // Store sum for normalization
            output[sz * sz] = sum;
        }

        // Synchronize before normalization
        __syncthreads();

        // Normalize using precomputed sum
        double sum = output[sz * sz];
        output[linearIdx] /= sum;
    }
}

__global__ void getGaussian(double *output, double sigma) {
    int sz = ceil(sigma * 3) * 2 + 1;
    if (threadIdx.y * sz + threadIdx.x >= sz * sz) return;  // Boundary check

    generate2DGaussian(output, sigma, sz, false);
}