#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void generate2DGaussian(double * output, double sigma, int sz, bool normalize) {

    // Thread index handling for better readability
    const int colIdx = threadIdx.x;
    const int rowIdx = threadIdx.y;
    int linearIdx = rowIdx*sz + colIdx;

    // Pre-compute constants and distances
    double invTwoSigmaSq = 1.0 / (2.0 * sigma * sigma);
    int distx = colIdx - sz / 2;
    int disty = rowIdx - sz / 2;

    // Calculate Gaussian value
    output[linearIdx] = exp(-(distx * distx + disty * disty) * invTwoSigmaSq);

    if(normalize) {
        // Synchronize threads before normalization
        __syncthreads();

        // Use a local variable to accumulate the sum
        double sum = 0.0;
        for(int i = 0; i < sz*sz; i++) {
            sum += output[i];
        }

        // Normalize values
        __syncthreads(); // Ensure sum calculation is complete
        output[linearIdx] /= sum;
    }
}

__global__ void getDoG(double * output, double sigma, double sigmaratio) {

    // Calculate filter size
    int sz = ceil(sigma*3) * 2 + 1;

    // Calculate linear index
    int linearIdx = threadIdx.y * sz + threadIdx.x;
    if(linearIdx >= sz * sz) return;

    // Shared memory for Gaussian filters
    extern __shared__ double sharedMemory[];
    double* g1 = sharedMemory;
    double* g2 = sharedMemory + 900;

    // Generate Gaussians in shared memory
    generate2DGaussian(g1, sigma, sz, true);
    generate2DGaussian(g2, sigma * sigmaratio, sz, true);

    // Synchronize threads before applying DoG
    __syncthreads();

    // Compute Difference of Gaussians
    output[linearIdx] = g2[linearIdx] - g1[linearIdx];
}