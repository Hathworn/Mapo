#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__  void simple_add(float* a, float* b, int n)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop to handle larger data sets by utilizing more threads effectively
    for (; i < n; i += blockDim.x * gridDim.x) {
        a[i] = a[i] + b[i];
    }
}