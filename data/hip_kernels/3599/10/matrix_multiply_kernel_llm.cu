#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    // Cache thread's index and reduce redundant calculations
    long i = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure work is done only for valid threads
    if (i < dim_mn) {
        double out = 0.0;  // Use optimized initialization
        // Loop unrolling could be applied for better performance but is skipped for simplicity
        for (long j = 0; j < dim_mn; j++) {
            out += matrix[i * dim_mn + j] * vector_in[j];
        }
        vector_out[i] = out;
    }
}