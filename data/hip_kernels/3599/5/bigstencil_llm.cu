#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bigstencil(int* in, int* out) {
    // Calculate the global thread index more efficiently using a built-in function
    int i = __mul24(blockIdx.x, blockDim.x) + threadIdx.x;

    // Unroll loop for better throughput (assuming input size allows for such unrolling)
    #pragma unroll
    for (int offset = 0; offset < 2; ++offset) {
        out[i + offset] = in[i + offset] + 2;
    }
}