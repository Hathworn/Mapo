#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];

    int tid = threadIdx.x;
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Load input into shared memory
    tmp[tid] = input[index];
    __syncthreads();

    // Unroll the loop to reduce instructions
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            tmp[tid] += tmp[tid + stride];
        }
        __syncthreads();
    }

    // Write the reduced value for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = tmp[0];
    }
}