#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Emptied global kernel, as it contains no operations
__global__ void null_kernel() {
    // Nothing to optimize here, kernel does nothing
};