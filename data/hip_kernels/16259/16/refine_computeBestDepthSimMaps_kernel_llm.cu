#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float refineDepthSubPixel(const float3& depths, const float3& sims)
{
    float outDepth = -1.0f;

    float simM1 = (sims.x + 1.0f) * 0.5f;
    float simP1 = (sims.z + 1.0f) * 0.5f;
    float sim1 = (sims.y + 1.0f) * 0.5f;

    if ((simM1 > sim1) && (simP1 > sim1))
    {
        float dispStep = -((simP1 - simM1) / (2.0f * (simP1 + simM1 - 2.0f * sim1)));
        float floatDepthM1 = depths.x;
        float floatDepthP1 = depths.z;
        float b = (floatDepthP1 + floatDepthM1) * 0.5f;
        float a = b - floatDepthM1;
        outDepth = a * dispStep + b;
    }

    return outDepth;
}

__global__ void refine_computeBestDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float3* isims, int isims_p, float3* idpts, int idpts_p, int width, int height, float simThr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        float3 depths = idpts[y * idpts_p + x];
        float3 sims = isims[y * isims_p + x];

        // Choosing the best depth and similarity
        float outDepth = (sims.x < sims.y) ? depths.x : depths.y;
        float outSim = (sims.x < sims.y) ? sims.x : sims.y;
        if (sims.z < outSim) {
            outDepth = depths.z;
            outSim = sims.z;
        }

        // Refine depth if possible
        float refinedDepth = refineDepthSubPixel(depths, sims);
        if (refinedDepth > 0.0f) {
            outDepth = refinedDepth;
        }

        // Update result arrays
        osim[y * osim_p + x] = (outSim < simThr ? outSim : 1.0f);
        odpt[y * odpt_p + x] = (outSim < simThr ? outDepth : -1.0f);
    }
}