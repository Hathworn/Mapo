#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generateGaussian_kernel(float* og, float delta, int radius)
{
    int x = threadIdx.x - radius;
    float deltaSquared = 2.0f * delta * delta; // Precompute constant part
    int threadIdxX = threadIdx.x;
    // Use precomputed constant in __expf
    og[threadIdxX] = __expf(-(x * x) / deltaSquared);
}