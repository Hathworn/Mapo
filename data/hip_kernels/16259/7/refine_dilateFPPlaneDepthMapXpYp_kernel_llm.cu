#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_dilateFPPlaneDepthMapXpYp_kernel(float* fpPlaneDepthMap, int fpPlaneDepthMap_p, float* maskMap, int maskMap_p, int width, int height, int xp, int yp, float fpPlaneDepth)
{
    // Compute global thread coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if coordinates are within bounds after translating by xp and yp
    int newX = x + xp;
    int newY = y + yp;
    if (newX >= 0 && newY >= 0 && newX < width && newY < height && x < width && y < height)
    {
        float depth = maskMap[y * maskMap_p + x];
        // Set fpPlaneDepthMap value if depth is greater than 0
        if (depth > 0.0f)
        {
            fpPlaneDepthMap[newY * fpPlaneDepthMap_p + newX] = fpPlaneDepth;
        }
    }
}