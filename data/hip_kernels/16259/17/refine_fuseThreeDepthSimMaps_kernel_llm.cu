#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_fuseThreeDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float* isimLst, int isimLst_p, float* idptLst, int idptLst_p, float* isimAct, int isimAct_p, float* idptAct, int idptAct_p, int width, int height, float simThr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < width) && (y < height))
    {
        // Load data from global to registers
        float dpt0 = odpt[y * odpt_p + x];
        float dpt1 = idptLst[y * idptLst_p + x];
        float dpt2 = idptAct[y * idptAct_p + x];

        float sim0 = osim[y * osim_p + x];
        float sim1 = isimLst[y * isimLst_p + x];
        float sim2 = isimAct[y * isimAct_p + x];

        // Reduce branch divergence by using select statements
        float outDpt = (sim1 < sim0 ? dpt1 : dpt0);
        float outSim = (sim1 < sim0 ? sim1 : sim0);
        outDpt = (sim2 < outSim ? dpt2 : outDpt);
        outSim = (sim2 < outSim ? sim2 : outSim);

        // Write results back to global memory
        osim[y * osim_p + x] = outSim;
        odpt[y * odpt_p + x] = outDpt;
    }
}