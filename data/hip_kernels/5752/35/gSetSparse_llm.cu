#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gSetSparse(float* out, const size_t* indices, const float* values, int length) {
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within the length
    if (index < length) {
        // Directly store value using calculated index
        out[indices[index]] = values[index];
    }
}