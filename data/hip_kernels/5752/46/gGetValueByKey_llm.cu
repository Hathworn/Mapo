#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gGetValueByKey(float* d_in, float* d_out, int* indeces, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Compute thread ID
    if(tid < n) {
        int index = __ldg(&indeces[tid]);  // Use __ldg for better memory load performance
        d_out[tid] = d_in[index]; // Retrieve value by index
    }
}