#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gScatterUpdate(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    // Use shared memory to reduce global memory access for sparse indices within a block
    extern __shared__ int sharedSparseIndices[];
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < sparseSize) {
        sharedSparseIndices[threadIdx.x] = sparseIndices[idx];
        __syncthreads(); // Synchronize to ensure shared memory is populated

        int sparseIdx = sharedSparseIndices[threadIdx.x];
        int targetIdx = sparseIdx + offset;
        
        // Perform boundary check before updating dense data
        if(targetIdx >= 0 && targetIdx < denseSize) {
            denseData[targetIdx] = sparseData[idx];
        }
    }
}