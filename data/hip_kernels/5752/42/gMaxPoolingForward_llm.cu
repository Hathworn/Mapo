#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gMaxPoolingForward(float* out, int outRows, int outCols, float* in, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Early exit if tid is out of bounds
    if (tid >= outRows * outCols) return;

    int rowId = tid / outCols; // Calculate correct rowId
    int colId = tid % outCols; // Calculate correct colId

    float* b = in + (rowId * inCols) + (colId * width);
    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;

    // Adjust width for the last column
    if (colId == outCols - 1) {
        width = lastWidth;
    }

    // Initialize currentMax with negative infinity to handle negative values
    float currentMax = -FLT_MAX;
    for (int i = 0; i < width; ++i) {
        float val = b[i] * localMask[i];
        if (val > currentMax) {
            currentMax = val;
        }
    }

    out[rowId * outCols + colId] = currentMax; // Correct indexing for output
}