#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gMaxPoolingBackward(float* adj, int adjRows, int adjCols, float* in, float* adjIn, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Early exit if thread ID is beyond the bounds
    if(tid >= adjRows * adjCols)
        return;

    // Calculate row and column indices
    int rowId = tid / adjCols; // Correcting column stride to adjCols
    int colId = tid % adjCols; // Correcting column stride to adjCols

    float* b = in + rowId * inCols + colId * width;

    // Adjust width for the last segment
    if(colId == adjCols - 1) { // Correct the comparison to adjCols
        width = lastWidth;
    }

    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;
    
    // Find max index more efficiently
    size_t currentMaxIdx = 0;
    float currentMaxValue = b[0] * localMask[0]; // Start comparison from the first element
    
    for(int i = 1; i < width; ++i) {
        float maskedValue = b[i] * localMask[i];
        if(maskedValue > currentMaxValue) {
            currentMaxIdx = i;
            currentMaxValue = maskedValue;
        }
    }

    // Update the corresponding index in adjIn
    atomicAdd(&adjIn[rowId * inCols + colId * width + currentMaxIdx], adj[rowId * adjCols + colId]); // Use atomic add for potential race conditions
}