#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gGather(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimization: Use grid-stride loop for better load balancing
    for (; idx < sparseSize; idx += gridDim.x * blockDim.x) {
        int sparseIdx = sparseIndices[idx] + offset;
        
        // Ensure sparseIdx is within bounds before accessing
        if (sparseIdx >= 0 && sparseIdx < denseSize) {
            sparseData[idx] = denseData[sparseIdx];
        }
    }
}