#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gScatterAdd(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    // Use shared memory for indices to reduce global memory access
    __shared__ int sharedSparseIndices[1024];

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Check bounds and load sparse indices into shared memory
    if (idx < sparseSize) {
        sharedSparseIndices[threadIdx.x] = sparseIndices[idx];
    }
    __syncthreads();

    // Ensure only threads within bounds perform addition
    if (idx < sparseSize) {
        int index = sharedSparseIndices[threadIdx.x];
        if (index >= -offset && index + offset < denseSize) {
            // Use atomic operation to avoid race conditions in parallel modification
            atomicAdd(&denseData[index + offset], sparseData[idx]);
        }
    }
}