#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void assignColIds(int* colIds, const int* colOffsets) {
    int myId = blockIdx.x;
    int start = colOffsets[myId];
    int end = colOffsets[myId + 1];
    int threadId = threadIdx.x;
    int blockDimSize = blockDim.x;

    // Use shared memory for faster access
    extern __shared__ int sharedColOffsets[];

    if (threadId < 2) {
        sharedColOffsets[threadId] = colOffsets[myId + threadId];
    }

    __syncthreads();

    start = sharedColOffsets[0];
    end = sharedColOffsets[1];

    for (int id = start + threadId; id < end; id += blockDimSize) {
        colIds[id] = myId;
    }
}