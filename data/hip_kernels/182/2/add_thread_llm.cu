#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_thread(int *a, int *b, int *c) {
    // Calculate global index for the thread within the grid
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform addition if within bounds
    if (index < blockDim.x) {
        c[index] = a[index] + b[index];
    }
}