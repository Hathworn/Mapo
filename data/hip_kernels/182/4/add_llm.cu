#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    // Calculate unique thread index within the grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform vector addition in parallel, avoid race condition
    if (idx == 0) {
        *c = *a + *b;
    }
}