#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot(int *a, int *b, int *c) {

    __shared__ int prod[THREADS_PER_BLOCK]; // Shared memory
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    prod[threadIdx.x] = a[index] * b[index];

    __syncthreads();  // Threads synchronization

    // Parallel reduction within block
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if (threadIdx.x % (2 * stride) == 0) {
            prod[threadIdx.x] += prod[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Atomic addition to global result
    if (threadIdx.x == 0) {
        atomicAdd(c, prod[0]); // Blocks reduction
    }
}