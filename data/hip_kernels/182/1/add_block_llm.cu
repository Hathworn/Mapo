#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_block(int *a, int *b, int *c) {
    // Use shared memory to optimize memory access within a block
    __shared__ int shared_a;
    __shared__ int shared_b;
    
    shared_a = a[blockIdx.x];
    shared_b = b[blockIdx.x];
    
    __syncthreads();  // Ensure all threads have loaded data
    
    // Perform the addition
    c[blockIdx.x] = shared_a + shared_b;
}