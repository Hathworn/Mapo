#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 128*256
#define THREADS_PER_BLOCK 256
#define N_BLOCKS N/THREADS_PER_BLOCK

// Optimize kernel to ensure it's executed only within valid index range
__global__ void add(int *a, int *b, int *c) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) { // Check to avoid out-of-bounds access
        c[index] = a[index] + b[index];
    }
}