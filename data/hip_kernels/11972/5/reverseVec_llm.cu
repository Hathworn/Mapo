#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void reverseVec(int n, float *a, float *b)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is in bounds
    if (i < n)
    {
        // Reverse copy from a to b
        b[n - 1 - i] = a[i];
    }
}