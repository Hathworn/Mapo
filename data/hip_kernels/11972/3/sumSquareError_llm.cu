#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void sumSquareError (int nBatch, int rbs, int rScale, int nCoeff, float *DA, float *CA, float *EA, float *SA)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nBatch)
    {
        const int daOffset = i * rbs * rScale * nCoeff;
        const int caOffset = i * nCoeff;
        const int eaOffset = i * rbs * rScale;

        float sum = 0.0f; // Local variable for sum square error
        for(int j = 0; j < rbs * rScale ; j++)
        {
            float fx = 0.0f;
            for(int k = 0; k < nCoeff ; k++)
            {
                fx += DA[daOffset + j + rbs * rScale * k] * CA[caOffset + k];
            }
            float error = EA[eaOffset + j] - fx;
            sum += error * error; // Accumulate sum of squared errors
        }
        SA[i] = sum; // Write back to global memory only once
    }
}