#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void limitCoeff (int nBatch, int rbs, int rScale, float maxCoeff, float *DA, float *RA, float *CA)
{
    int taskIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (taskIdx < nBatch)
    {
        int i = taskIdx;
        int nCoeff = 2;

        int daOffset = i * rbs * rScale * nCoeff;
        int raOffset = i * rbs * rScale;
        int caOffset = i * nCoeff;

        // Use shared memory for coefficients
        __shared__ float sharedCA[2];

        sharedCA[0] = CA[caOffset];
        sharedCA[1] = CA[caOffset + 1];

        // Check and adjust coefficients using atomic operations
        if (sharedCA[1] > maxCoeff || sharedCA[1] < -maxCoeff) {
            sharedCA[1] = sharedCA[1] > maxCoeff ? maxCoeff : (sharedCA[1] < -maxCoeff ? -maxCoeff : sharedCA[1]);

            float suma = 0.0f;
            float sumb = 0.0f;

            // Use loop unrolling for efficient computation
            for (int j = 0; j < rbs * rScale; j += 4) {
                suma += DA[daOffset + j] + DA[daOffset + j + 1] + DA[daOffset + j + 2] + DA[daOffset + j + 3];
                sumb += RA[raOffset + j] + RA[raOffset + j + 1] + RA[raOffset + j + 2] + RA[raOffset + j + 3];
            }

            // Use atomic operation to update CA[caOffset]
            atomicExch(&CA[caOffset], (sumb - sharedCA[1] * suma) / (rbs * rScale));
            // Update the second coefficient back to global memory
            atomicExch(&CA[caOffset + 1], sharedCA[1]);
        }
    }
}