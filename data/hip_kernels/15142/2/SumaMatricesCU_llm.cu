#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SumaMatricesCU(int* A, int* B, int* C, int width) {
    // Calculate global index for thread
    int idx = blockIdx.y * blockDim.y * width + blockIdx.x * blockDim.x + threadIdx.y * width + threadIdx.x;
    
    // Ensure index is within matrix bounds
    if (idx < width * width) {
        C[idx] = A[idx] + B[idx]; // Perform addition
    }
}