#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZE_thread 1024

__global__ void VectorAdd(int *A, int *B, int *C, int n)
{
    // Use a more efficient block and thread calculation
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds before performing operations
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}