#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;

    // Use shared memory for coalesced access; assume enough shared memory
    extern __shared__ float shared_A[];
    float2 *shared_A_val = reinterpret_cast<float2 *>(shared_A);

    // Load data into shared memory
    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        shared_A_val[idx] = reinterpret_cast<float2 *>(Ar)[idx];
    }
    __syncthreads();

    // Compute power spectrum
    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        if (idx == 0) continue; // Ignore special case

        float2 val = shared_A_val[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }

    __syncthreads();

    // Handle special case for the first and last elements
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}