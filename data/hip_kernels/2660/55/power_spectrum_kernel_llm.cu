#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    const float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;

    // Use shared memory for intermediate values
    extern __shared__ float shared_mem[];
    float2 *shared_val = reinterpret_cast<float2*>(shared_mem);

    // Load data into shared memory
    if (thread_id < half_length) {
        shared_val[thread_id] = reinterpret_cast<const float2 *>(Ar)[thread_id];
    }
    __syncthreads();

    // Compute power spectrum using shared memory
    for (int idx = thread_id; idx < half_length; idx += CU1DBLOCK) {
        if (idx == 0) continue;

        float2 val = shared_val[idx];
        float ret = val.x * val.x + val.y * val.y;
        if (use_power) {
            Aw[idx] = ret;
        } else {
            Aw[idx] = sqrtf(ret);
        }
    }

    // Handle special case, only one thread operates
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabs(real);
            Aw[half_length] = fabs(im);
        }
    }
}
```
