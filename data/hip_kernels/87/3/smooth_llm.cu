#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth(float * v_new, const float * v) {
    // Calculate global thread index using a more standard approach
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Assuming a proper grid size is set where myIdx < numElements
    float myElt = v[myIdx];
    float myLeftElt = (myIdx > 0) ? v[myIdx - 1] : myElt; // Handle left boundary
    float myRightElt = (myIdx < blockDim.x * gridDim.x - 1) ? v[myIdx + 1] : myElt; // Handle right boundary

    // Perform smoothing operation
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
}