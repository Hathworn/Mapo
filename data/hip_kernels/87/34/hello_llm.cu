#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use thread index for more precise identification
    printf("Hello world! I'm thread %d in block %d\n", threadIdx.x, blockIdx.x);
}