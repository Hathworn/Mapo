#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void naive_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Calculate global thread ID
    int myId = threadIdx.x + blockDim.x * blockIdx.x;

    // Shared memory for bins to reduce global memory contention
    extern __shared__ int shared_bins[];

    // Initialize shared memory bins
    for (int i = threadIdx.x; i < BIN_COUNT; i += blockDim.x) {
        shared_bins[i] = 0;
    }
    __syncthreads();

    // Calculate bin for the current item
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;

    // Atomic addition to update the shared bin
    atomicAdd(&shared_bins[myBin], 1);
    __syncthreads();

    // Transfer shared memory bins to global memory
    for (int i = threadIdx.x; i < BIN_COUNT; i += blockDim.x) {
        atomicAdd(&d_bins[i], shared_bins[i]);
    }
}