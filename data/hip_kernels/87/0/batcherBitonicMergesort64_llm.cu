#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// http://en.wikipedia.org/wiki/Bitonic_sort

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid  = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();
    
    for (int stage = 0; stage <= 5; stage++)
    {
        for (int substage = stage; substage >= 0; substage--)
        {
            // Define the comparator distance
            int comparatorDistance = 1 << substage;

            // Calculate the indices to compare
            int index = tid ^ comparatorDistance;

            // Only perform comparisons within bounds and when index > tid
            if (index > tid)
            {
                // Ascend for even indexed sequences and descend for odd indexed
                if ((tid & (1 << stage)) == 0)
                {
                    // Ascending order
                    if (sdata[tid] > sdata[index])
                    {
                        // Swap if out of order
                        float temp = sdata[tid];
                        sdata[tid] = sdata[index];
                        sdata[index] = temp;
                    }
                }
                else
                {
                    // Descending order
                    if (sdata[tid] < sdata[index])
                    {
                        // Swap if out of order
                        float temp = sdata[tid];
                        sdata[tid] = sdata[index];
                        sdata[index] = temp;
                    }
                }
            }
            __syncthreads(); // Ensure threads are synchronized between stages
        }
    }
    
    d_out[tid] = sdata[tid];
}