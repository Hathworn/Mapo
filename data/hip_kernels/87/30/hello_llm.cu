#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Load thread index into a register to minimize repeated memory accesses
    int tid = threadIdx.x;

    // Reduce the frequency of printf by using a conditional statement
    if (tid < 32)  // Example: only first warp will print
    {
        printf("Hello world! I'm thread %d\n", tid);
    }
}