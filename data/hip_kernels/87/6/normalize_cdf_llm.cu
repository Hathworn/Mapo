#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    // Calculate normalization constant outside of the loop
    const float normalization_constant = 1.f / d_input_cdf[n - 1];
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_index_1d < n)
    {
        // Directly compute and assign output value
        d_output_cdf[global_index_1d] = d_input_cdf[global_index_1d] * normalization_constant;
    }
}