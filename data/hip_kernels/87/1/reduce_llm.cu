#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
    // Store thread's input value in shared memory
    s[threadIdx.x] = p;
    __syncthreads(); // Ensure all threads have written to shared memory

    // Perform warp-level reduction using shared memory
    for (int stride = 16; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            s[threadIdx.x] += s[threadIdx.x + stride];
        }
        __syncthreads(); // Synchronize to ensure all additions are done
    }

    // Result is stored in s[0]
    return s[0];
}

__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];

    // Compute the sum in shared memory reduction
    unsigned int sr = shared_reduce(p, s);

    // Only the first thread writes the result to the output
    if (t == 0) {
        *d_out_shared = sr;
    }
}