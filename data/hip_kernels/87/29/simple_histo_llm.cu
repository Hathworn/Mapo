#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;

    // Ensuring only valid global indices access memory
    if (myId < BIN_COUNT)
    {
        int myItem = d_in[myId];
        int myBin = myItem % BIN_COUNT;

        // Optimized via atomicAdd for concurrent execution safety
        atomicAdd(&(d_bins[myBin]), 1);
    }
}