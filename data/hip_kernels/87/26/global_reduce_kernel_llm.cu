#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
    extern __shared__ float sdata[]; // Allocate shared memory
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // Load data into shared memory
    sdata[tid] = d_in[myId];
    __syncthreads(); // Ensure all data is loaded

    // Perform reduction using shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Sync threads to ensure previous stage is done
    }

    // Write the result for this block to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}