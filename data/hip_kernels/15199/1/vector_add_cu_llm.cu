#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vector_add_cu(float *out, float *a, float *b, int n) {
    // Calculate global index using block and thread indices
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure no excess iterations by checking index against n
    if (index < n) {
        out[index] = a[index] + b[index];
    }
}