#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vector_add_cu(float *out, float *a, float *b, int n) {
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index is within bounds
    if (idx < n) {
        out[idx] = a[idx] + b[idx];
    }
}