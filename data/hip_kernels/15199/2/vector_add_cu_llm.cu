#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vector_add_cu(float *out, float *a, float *b, int n) {
    // Calculate the global index for this thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index is within bounds
    if (i < n) {
        out[i] = a[i] + b[i];
    }
}