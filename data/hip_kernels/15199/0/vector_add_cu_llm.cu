#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vector_add_cu(float *out, float *a, float *b, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if (idx < n) { // Ensure index is within bounds
        out[idx] = a[idx] + b[idx]; // Vector addition
    }
}