#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    extern __shared__ float shared_grad_out[];
    
    int batch_index = blockIdx.x;
    idx += m * nsample * batch_index;
    grad_out += m * nsample * c * batch_index;
    grad_points += n * c * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;
    int j_stride = index + blockIdx.y * stride; // Spread by blockIdx.y for better parallelism

    for (int j = j_stride; j < m; j += stride * gridDim.y) { // Grid-stride loop
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            for (int l = threadIdx.z; l < c; l += blockDim.z) { // Spread computation across z-dimension
                // Load grad_out into shared memory
                shared_grad_out[threadIdx.x * nsample * c + k * c + l] = grad_out[j * nsample * c + k * c + l];
                __syncthreads();

                // Use shared memory to reduce global memory access latency
                atomicAdd(&grad_points[ii * c + l], shared_grad_out[threadIdx.x * nsample * c + k * c + l]);
            }
        }
    }
}