#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    // Calculate batch offset
    int batch_index = blockIdx.x;
    idx += m * nsample * batch_index;
    grad_out += m * nsample * c * batch_index;
    grad_points += n * c * batch_index;

    // Calculate the thread-specific starting index
    int index = threadIdx.x;
    int stride = blockDim.x;

    // Loop over m with stride for better load balancing
    for (int j = index; j < m; j += stride) {
        // Unroll the nsample loop for better warp utilization
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            // Unroll the c loop for better warp utilization
            #pragma unroll
            for (int l = 0; l < c; ++l) {
                // Use atomic add to avoid race condition
                atomicAdd(&grad_points[ii * c + l], grad_out[j * nsample * c + k * c + l]);
            }
        }
    }
}