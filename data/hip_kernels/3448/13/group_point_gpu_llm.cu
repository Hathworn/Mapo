#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n * c * batch_index;
    idx += m * nsample * batch_index;
    out += m * nsample * c * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;
    int j = index;

    // Iterate over potentially large dimensions in parallel
    while (j < m) {
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            #pragma unroll  // Unroll the loop for better memory access patterns
            for (int l = 0; l < c; ++l) {
                out[j * nsample * c + k * c + l] = points[ii * c + l];
            }
        }
        j += stride;
    }
}