#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    // Calculate global thread ID
    int batch_idx = blockIdx.x;
    int point_idx = threadIdx.x;

    if (batch_idx < b && point_idx < m * nsample) {
        // Calculate the sample index within the batch
        int sample_idx = point_idx;
        
        // Get the point index from indices
        int ii = idx[sample_idx];
        
        // Use a loop to accumulate the gradients for the channels
        for (int l = 0; l < c; ++l) {
            atomicAdd(&grad_points[ii * c + l], grad_out[sample_idx * c + l]);
        }
    }
}