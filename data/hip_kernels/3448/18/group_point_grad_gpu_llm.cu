#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_idx = blockIdx.x;
    int point_idx = threadIdx.x + blockIdx.y * blockDim.x;

    if (point_idx < m) {
        idx += batch_idx * m * nsample;
        grad_out += batch_idx * m * nsample * c;
        grad_points += batch_idx * n * c;

        for (int k = 0; k < nsample; ++k) {
            int ii = idx[point_idx * nsample + k];
            for (int l = 0; l < c; ++l) {
                atomicAdd(&grad_points[ii * c + l], grad_out[point_idx * nsample * c + k * c + l]);
                // Optimized with atomicAdd to prevent race conditions and improve performance
            }
        }
    }
}