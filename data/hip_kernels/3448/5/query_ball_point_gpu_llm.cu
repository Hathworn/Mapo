#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
  // Get the block and thread index
  int batch_idx = blockIdx.x;
  int point_idx = blockIdx.y * blockDim.x + threadIdx.x;

  // Ensure within bounds
  if (batch_idx < b && point_idx < m) {
    float x2 = xyz2[point_idx * 3 + 0];
    float y2 = xyz2[point_idx * 3 + 1];
    float z2 = xyz2[point_idx * 3 + 2];
    int cnt = 0;
    for (int k=0; k<n; ++k) {
      if (cnt == nsample) break; // only pick the FIRST nsample points in the ball
      float x1 = xyz1[k * 3 + 0];
      float y1 = xyz1[k * 3 + 1];
      float z1 = xyz1[k * 3 + 2];
      float d = max(sqrtf((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1)), 1e-20f);
      if (d < radius) {
        if (cnt == 0) {
          // set ALL indices to k, to ensure we have valid (repeating) indices
          for (int l = 0; l < nsample; ++l)
            idx[point_idx * nsample + l] = k;
        }
        idx[point_idx * nsample + cnt] = k;
        cnt += 1;
      }
    }
  }
}