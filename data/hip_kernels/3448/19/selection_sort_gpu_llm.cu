#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, float *dist, int *idx, float *val) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    idx += m * k * batch_index;
    val += m * k * batch_index;
    
    int index = threadIdx.x;
    int stride = blockDim.x;
    
    float *p_dist;
    for (int j = index; j < m; j += stride) {
        p_dist = dist + j * n;
        
        // Use shared memory for better cache performance
        __shared__ float shared_dist[1024];
        
        // Copy data to shared memory for current thread's portion
        for (int s = 0; s < n; s += stride) {
            if (s + index < n) {
                shared_dist[s + index] = p_dist[s + index];
            }
        }
        __syncthreads();
        
        // Selection sort for the first k elements
        for (int s = 0; s < k; ++s) {
            int min = s;
            // Find the min
            for (int t = s + 1; t < n; ++t) {
                if (shared_dist[t] < shared_dist[min]) {
                    min = t;
                }
            }
            // Update idx and val
            idx[j * n + s] = min;
            val[j * n + s] = shared_dist[min];
            // Swap min-th and s-th element
            float tmp = shared_dist[min];
            shared_dist[min] = shared_dist[s];
            shared_dist[s] = tmp;
        }
    }
}