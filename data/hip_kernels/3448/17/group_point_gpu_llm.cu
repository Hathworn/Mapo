#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Macro definitions would go here if there are any

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    int thread_index = threadIdx.x;

    points += n * c * batch_index;  // Move points pointer to the current batch
    idx += m * nsample * batch_index;  // Move idx pointer to the current batch
    out += m * nsample * c * batch_index;  // Move out pointer to the current batch

    // Optimize loop by unrolling
    for (int j = thread_index; j < m; j += blockDim.x) {
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            #pragma unroll 4  // Unroll the innermost loop for better performance
            for (int l = 0; l < c; ++l) {
                out[j * nsample * c + k * c + l] = points[ii * c + l];
            }
        }
    }
}