#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    outi += m * n * batch_index;
    out  += m * n * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Optimize index calculation and use shared memory for temporary storage
    extern __shared__ float shared_out[];
    extern __shared__ int shared_outi[];

    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            shared_out[j * n + s] = dist[j * n + s];
            shared_outi[j * n + s] = s;
        }
    }
    __syncthreads();

    float *p_dist;
    for (int j = index; j < m; j += stride) {
        p_dist = shared_out + j * n;
        // Selection sort for the first k elements
        for (int s = 0; s < k; ++s) {
            int min = s;
            // Find the minimum
            for (int t = s + 1; t < n; ++t) {
                if (p_dist[t] < p_dist[min]) {
                    min = t;
                }
            }
            // Swap min-th and i-th element
            if (min != s) {
                float tmp = p_dist[min];
                p_dist[min] = p_dist[s];
                p_dist[s] = tmp;
                int tmpi = shared_outi[j * n + min];
                shared_outi[j * n + min] = shared_outi[j * n + s];
                shared_outi[j * n + s] = tmpi;
            }
        }
    }
    __syncthreads();

    // Write back to global memory
    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            out[j * n + s] = shared_out[j * n + s];
            outi[j * n + s] = shared_outi[j * n + s];
        }
    }
}