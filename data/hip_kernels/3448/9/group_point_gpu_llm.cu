#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n * c * batch_index;
    idx += m * nsample * batch_index;
    out += m * nsample * c * batch_index;

    int index = threadIdx.x + blockIdx.y * blockDim.x;  // Use blockIdx.y for additional parallelism
    int stride = blockDim.x * gridDim.y;  // Adjust stride for additional y-dim blocks

    // Use deeper nesting for coalesced memory access on out and points
    for (int j = index; j < m; j += stride) { 
        for (int l = 0; l < c; ++l) {  
            for (int k = 0; k < nsample; ++k) {
                int ii = idx[j * nsample + k];
                out[j * nsample * c + k * c + l] = points[ii * c + l];
            }
        }
    }
}