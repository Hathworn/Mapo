#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void jackpot_compactTest_gpu_ADD(uint32_t *data, uint32_t *partial_sums, int len)
{
    __shared__ uint32_t buf;
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread accesses within bounds
    if (id >= len) return;

    // Load from global memory to shared memory by first thread
    if (threadIdx.x == 0) {
        buf = partial_sums[blockIdx.x];
    }

    __syncthreads();

    // Add shared memory value
    data[id] += buf;
}