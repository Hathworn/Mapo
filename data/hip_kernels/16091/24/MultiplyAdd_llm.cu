#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate global thread index for x and y
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Flatten index for 1D memory access
    int p = y * width + x;

    // Check bounds and perform computation
    if (x < width && y < height)
    {
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
}