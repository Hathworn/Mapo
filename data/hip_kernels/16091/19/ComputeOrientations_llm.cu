#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h)
{
    __shared__ float data[16 * 15];
    __shared__ float hist[32 * 13];
    __shared__ float gauss[16];

    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    
    // Initialize histogram bins to zero
    for (int i = 0; i < 13; i++) {
        hist[i * 32 + tx] = 0.0f;
    }
    __syncthreads();

    // Precompute Gaussian weights
    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15)
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));
    
    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;
    int px = xp & 15;
    int x = tx - px;

    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = xp + x;
        int yi = yp + y;

        // Clamp coordinates inside image boundaries
        xi = max(0, min(xi, w - 1));
        yi = max(0, min(yi, h - 1));

        if (x >= 0 && x < 15) 
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = 16.0f * atan2f(dy, dx) / 3.1416f + 16.5f;

            if (bin == 32)
                bin = 0;
            
            float grad = sqrtf(dx * dx + dy * dy);
            hist[32 * (x - 1) + bin] += grad * gauss[x] * gauss[y];
        }
    }
    __syncthreads();

    // Aggregation of histogram bins
    for (int offset = 8; offset >= 1; offset /= 2) {
        if (tx < 32) {
            for (int y = 0; y < (8 / offset); y++) {
                hist[y * 32 + tx] += hist[(y + offset) * 32 + tx];
            }
        }
        __syncthreads();
    }

    if (tx < 32)
        hist[tx] += hist[32 + tx];
    __syncthreads();

    // Apply smoothing to histogram
    if (tx == 0) {
        hist[32] = 6 * hist[0] + 4 * (hist[1] + hist[31]) + (hist[2] + hist[30]);
    } else if (tx == 1) {
        hist[33] = 6 * hist[1] + 4 * (hist[2] + hist[0]) + (hist[3] + hist[31]);
    } else if (tx >= 2 && tx <= 29) {
        hist[tx + 32] = 6 * hist[tx] + 4 * (hist[tx + 1] + hist[tx - 1]) +
                        (hist[tx + 2] + hist[tx - 2]);
    } else if (tx == 30) {
        hist[62] = 6 * hist[30] + 4 * (hist[31] + hist[29]) + (hist[0] + hist[28]);
    } else if (tx == 31) {
        hist[63] = 6 * hist[31] + 4 * (hist[0] + hist[30]) + (hist[1] + hist[29]);
    }
    __syncthreads();

    // Find maximum orientation peaks
    float v = hist[32 + tx];
    if (tx < 32) {
        hist[tx] = (v > hist[32 + ((tx + 1) & 31)] && v >= hist[32 + ((tx + 31) & 31)]) ? v : 0.0f;
    }
    __syncthreads();

    if (tx == 0) {
        float maxval1 = 0.0f, maxval2 = 0.0f;
        int i1 = -1, i2 = -1;

        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }

        float val1 = hist[32 + ((i1 + 1) & 31)];
        float val2 = hist[32 + ((i1 + 31) & 31)];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Orient[bx] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);

        if (maxval2 < 0.8f * maxval1)
            i2 = -1;

        if (i2 >= 0) {
            val1 = hist[32 + ((i2 + 1) & 31)];
            val2 = hist[32 + ((i2 + 31) & 31)];
            peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Orient[bx + maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        } else {
            d_Orient[bx + maxPts] = i2;
        }
    }
}