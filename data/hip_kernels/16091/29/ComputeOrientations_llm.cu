#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize histogram
    for (int i=0; i<13; i++) 
        hist[i*32+tx] = 0.0f;
    __syncthreads();

    // Precompute Gaussian
    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15)
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));
    __syncthreads();

    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;
    int px = xp & 15;
    int x = tx - px;

    // Load data into shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16*y + x;
        int xi = xp + x;
        int yi = yp + y;
        if (xi < 0) xi = 0;
        if (xi >= w) xi = w - 1;
        if (yi < 0) yi = 0;
        if (yi >= h) yi = h - 1;
        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Compute gradients and histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16*y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos+16] - data[memPos-16];
            float dx = data[memPos+1]  - data[memPos-1];
            float grad = sqrtf(dx * dx + dy * dy);
            int bin = __float2int_rn(16.0f * atan2f(dy, dx) / 3.1416f + 16.5f) % 32; // Optimized bin calculation
            hist[32*(x-1) + bin] += grad * gauss[x] * gauss[y];
        }
    }
    __syncthreads();

    // Reduce histogram
    for (int offset = 8; offset > 0; offset /= 2) {
        if (tx < offset * 32) 
            hist[tx] += hist[tx + offset * 32];
        __syncthreads();
    }
    
    // Smoothing the histogram
    float v = hist[tx + 32];
    hist[tx] = (v > hist[(tx+1)&31 + 32] && v >= hist[(tx+31)&31 + 32]) ? v : 0.0f;
    __syncthreads();

    // Find peaks of the histogram
    if (tx == 0) {
        float maxval1 = 0.0, maxval2 = 0.0;
        int i1 = -1, i2 = -1;
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }
        if (i1 >= 0) {
            float val1 = hist[(i1+1)&31 + 32];
            float val2 = hist[(i1+31)&31 + 32];
            float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
            d_Orient[bx] = 11.25f * ((peak < 0.0f) ? peak + 32.0f : peak);

            if (maxval2 < 0.8f * maxval1) i2 = -1;
            if (i2 >= 0) {
                val1 = hist[(i2+1)&31 + 32];
                val2 = hist[(i2+31)&31 + 32];
                peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
                d_Orient[bx + maxPts] = 11.25f * ((peak < 0.0f) ? peak + 32.0f : peak);
            } else {
                d_Orient[bx + maxPts] = i2;
            }
        }
    }
}