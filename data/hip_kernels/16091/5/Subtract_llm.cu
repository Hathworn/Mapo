#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SUBTRACT_W 16 // Example block width, adjust as needed
#define SUBTRACT_H 16 // Example block height, adjust as needed

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Align threadIdx and blockDim
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Align threadIdx and blockDim
    if (x < width && y < height) {
        int p = y * pitch + x; // Calculate index only if within bounds
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}
```
