#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;

    // Loop unrolling and using shared memory for minor performance gain
    extern __shared__ unsigned int sharedMinmax[];

    for (int y = 0; y < h / 32; y++) {
        for (int x = 0; x < w; x += 16) {
            sharedMinmax[tx] = minmax[y * w + x + tx];

            if (sharedMinmax[tx]) {
                unsigned int val = sharedMinmax[tx];
                #pragma unroll
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}