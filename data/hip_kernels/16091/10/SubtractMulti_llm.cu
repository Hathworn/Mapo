#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Calculate global indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z; // Use 3D grid for better utilization

    int sz = height * pitch;
    int p = z * sz + y * pitch + x;
    
    // Ensure in-bound checks
    if (x < width && y < height)
    {
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
    // Remove __syncthreads since no shared memory is used
}