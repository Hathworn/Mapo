#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int height) {
    // Data cache with reduced shared memory usage
    __shared__ float data1[3 * (MINMAX_SIZE + 2)];
    __shared__ float data2[3 * (MINMAX_SIZE + 2)];
    __shared__ float data3[3 * (MINMAX_SIZE + 2)];
    __shared__ float ymin2[(MINMAX_SIZE + 2)];
    __shared__ float ymax2[(MINMAX_SIZE + 2)];

    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_SIZE;
    const int xEndClamped = min(xStart + MINMAX_SIZE - 1, width - 1);
    int memWid = MINMAX_SIZE + 2;

    int memPos0 = (tx - WARP_SIZE + 1);
    int memPos1 = (tx - WARP_SIZE + 1);
    int yq = 0;
    unsigned int output = 0;

    for (int y = 0; y < 34; y++) {
        output >>= 1;
        int memPos = yq * memWid + (tx - WARP_SIZE + 1);
        int yp = 32 * blockIdx.y + y - 1;
        yp = max(yp, 0);
        yp = min(yp, height - 1);
        int readStart = yp * width;

        if (tx >= (WARP_SIZE - 1)) {
            int xReadPos = xStart + tx - WARP_SIZE;
            if (xReadPos >= 0 && xReadPos < width) {
                data1[memPos] = d_Data1[readStart + xReadPos];
                data2[memPos] = d_Data2[readStart + xReadPos];
                data3[memPos] = d_Data3[readStart + xReadPos];
            } else {
                data1[memPos] = 0;
                data2[memPos] = 0;
                data3[memPos] = 0;
            }
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;
        if (y > 1) {
            if (tx < memWid) {
                float min1 = fminf(data1[memPos0], data1[memPos2]);
                float min2 = fminf(fminf(data2[memPos0], data2[memPos1]), data2[memPos2]);
                float min3 = fminf(data3[memPos1], data3[memPos2]);

                float max1 = fmaxf(data1[memPos0], data1[memPos2]);
                float max2 = fmaxf(fmaxf(data2[memPos0], data2[memPos1]), data2[memPos2]);
                float max3 = fmaxf(data3[memPos1], data3[memPos2]);

                ymin2[tx] = fminf(min1, fminf(min2, min3));
                ymax2[tx] = fmaxf(max1, fmaxf(max2, max3));
            }
        }
        __syncthreads();

        if (y > 1 && tx < MINMAX_SIZE) {
            int xWritePos = xStart + tx;
            if (xWritePos <= xEndClamped) {
                float minv = fminf(fminf(ymin2[tx], ymin2[tx + 2]), data2[memPos2 + 1]);
                minv = fminf(minv, d_ConstantA[1]);
                float maxv = fmaxf(fmaxf(ymax2[tx], ymax2[tx + 2]), data2[memPos2 + 1]);
                maxv = fmaxf(maxv, d_ConstantA[0]);
                if (data2[memPos1 + 1] < minv || data2[memPos1 + 1] > maxv)
                    output |= 0x80000000;
            }
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }

    if (tx < MINMAX_SIZE) {
        int xWritePos = xStart + tx;
        if (xWritePos < width) {
            int writeStart = blockIdx.y * width + xWritePos;
            d_Result[writeStart] = output;
        }
    }
}