#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height) {
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    
    int tx = threadIdx.x;
    int x = blockIdx.x * 128 + tx;
    int y = blockIdx.y * 16;
    int p = y * width + x;
    
    // Load initial min/max values into shared memory
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        minvals[tx] = FLT_MAX;
        maxvals[tx] = -FLT_MAX;
    }
    
    // Collaborative loading of data by threads in the block
    for (int ty = 1; ty < 16; ty++) {
        p += width;
        if (x < width) {
            float val = d_Data[p];
            if (val < minvals[tx]) minvals[tx] = val;
            if (val > maxvals[tx]) maxvals[tx] = val;
        }
    }
    __syncthreads();
    
    // Parallel reduction for min/max values
    for (unsigned int s = warpSize / 2; s > 0; s >>= 1) {
        if (tx < s && tx + s < 128) {
            if (minvals[tx + s] < minvals[tx]) minvals[tx] = minvals[tx + s];
            if (maxvals[tx + s] > maxvals[tx]) maxvals[tx] = maxvals[tx + s];
        }
        __syncthreads();
    }
    
    // Write the results from shared to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}