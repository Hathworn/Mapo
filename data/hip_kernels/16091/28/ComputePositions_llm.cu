#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
    int i = blockIdx.x * POSBLK_SIZE + threadIdx.x; // Use standard operations instead of __mul24
    if (i >= numPts) return;

    int p = d_Ptrs[i];
    
    // Prefetch common memory accesses and avoid repeated indexing
    float val2_p = g_Data2[p];
    float val3_p = g_Data3[p];
    float val1_p = g_Data1[p];

    float val[7];
    val[0] = val2_p;
    val[1] = g_Data2[p - 1];
    val[2] = g_Data2[p + 1];
    
    float dx = 0.5f * (val[2] - val[1]);
    float dxx = 2.0f * val2_p - val[1] - val[2];
    
    val[3] = g_Data2[p - w];
    val[4] = g_Data2[p + w];
    
    float dy = 0.5f * (val[4] - val[3]);
    float dyy = 2.0f * val2_p - val[3] - val[4];
    
    val[5] = val3_p;
    val[6] = val1_p;
    
    float ds = 0.5f * (val1_p - val3_p);
    float dss = 2.0f * val2_p - val3_p - val1_p;
    
    float dxy = 0.25f * 
                (g_Data2[p + w + 1] + g_Data2[p - w - 1] - g_Data2[p - w + 1] - g_Data2[p + w - 1]);
    float dxs = 0.25f * 
                (g_Data3[p + 1] + g_Data1[p - 1] - g_Data1[p + 1] - g_Data3[p - 1]);
    float dys = 0.25f * 
                (g_Data3[p + w] + g_Data1[p - w] - g_Data3[p - w] - g_Data1[p + w]);

    // Calculate determinant components
    float idxx = dyy * dss - dys * dys;
    float idxy = dys * dxs - dxy * dss;
    float idxs = dxy * dys - dyy * dxs;
    float idyy = dxx * dss - dxs * dxs;
    float idys = dxy * dxs - dxx * dys;
    float idss = dxx * dyy - dxy * dxy;

    // Avoid recalculating inverses, reduced duplicate floats
    float det = idxx * dxx + idxy * dxy + idxs * dxs;
    float idet = 1.0f / det;
    
    float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
    float pdy = idet * (idxy * dx + idyy * dy + idys * ds);
    float pds = idet * (idxs * dx + idys * dy + idss * ds);
    
    // Inline compare and clamp using intrinsics, minimizing branching overhead
    if (pdx < -0.5f || pdx > 0.5f || pdy < -0.5f || pdy > 0.5f || pds < -0.5f || pds > 0.5f) {
        pdx = __fdividef(dx, dxx);
        pdy = __fdividef(dy, dyy);
        pds = __fdividef(ds, dss);
    }
    
    float dval = 0.5f * (dx * pdx + dy * pdy + ds * pds);
    d_Sift[i + 0 * maxPts] = (p % w) + pdx;
    d_Sift[i + 1 * maxPts] = (p / w) + pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val2_p + dval;
    
    float tra = dxx + dyy;
    det = dxx * dyy - dxy * dxy;
    d_Sift[i + 4 * maxPts] = __fdividef(tra * tra, det);
}