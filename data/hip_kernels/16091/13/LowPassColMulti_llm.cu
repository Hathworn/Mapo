#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassColMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVCOL_W*(CONVCOL_H + 2*RADIUS)];
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int block = blockIdx.x/(NUM_SCALES+3);
    const int scale = blockIdx.x - (NUM_SCALES+3)*block;
    const int miny = blockIdx.y*CONVCOL_H;
    const int maxy = min(miny + CONVCOL_H, height) - 1;
    const int totStart = miny - RADIUS;
    const int totEnd = maxy + RADIUS;
    const int colStart = block*CONVCOL_W + tx;
    const int colEnd = colStart + (height-1)*pitch;
    const int sStep = CONVCOL_W*CONVCOL_S;
    const int gStep = pitch*CONVCOL_S;
    float *kernel = d_Kernel + scale*16;
    const int size = pitch*height*scale;
    d_Result += size;
    d_Data += size;

    if (colStart < width) {
        float *sdata = data + ty*CONVCOL_W + tx;
        int gPos = colStart + (totStart + ty)*pitch;
        
        // Use loop unrolling for memory loading
        for (int y = totStart + ty; y <= totEnd; y += blockDim.y * 4) {
            sdata[0] = (y < 0) ? d_Data[colStart] : (y >= height) ? d_Data[colEnd] : d_Data[gPos];
            sdata += sStep;
            gPos += gStep;
            if (y + blockDim.y * 1 <= totEnd) {
                sdata[0] = (y + blockDim.y * 1 < 0) ? d_Data[colStart] : (y + blockDim.y * 1 >= height) ? d_Data[colEnd] : d_Data[gPos];
                sdata += sStep;
                gPos += gStep;
            }
            if (y + blockDim.y * 2 <= totEnd) {
                sdata[0] = (y + blockDim.y * 2 < 0) ? d_Data[colStart] : (y + blockDim.y * 2 >= height) ? d_Data[colEnd] : d_Data[gPos];
                sdata += sStep;
                gPos += gStep;
            }
            if (y + blockDim.y * 3 <= totEnd) {
                sdata[0] = (y + blockDim.y * 3 < 0) ? d_Data[colStart] : (y + blockDim.y * 3 >= height) ? d_Data[colEnd] : d_Data[gPos];
            }
        }
    }
    __syncthreads();

    if (colStart < width) {
        float *sdata = data + ty*CONVCOL_W + tx;
        int gPos = colStart + (miny + ty)*pitch;
        
        // Optimize computation loop using loop unrolling
        for (int y = miny + ty; y <= maxy; y += blockDim.y*4) {
            #pragma unroll
            for (int offset = 0; offset < 4 && y + offset * blockDim.y <= maxy; ++offset) {
                d_Result[gPos + offset * gStep] =
                    (sdata[0*CONVCOL_W] + sdata[8*CONVCOL_W])*kernel[0] +
                    (sdata[1*CONVCOL_W] + sdata[7*CONVCOL_W])*kernel[1] +
                    (sdata[2*CONVCOL_W] + sdata[6*CONVCOL_W])*kernel[2] +
                    (sdata[3*CONVCOL_W] + sdata[5*CONVCOL_W])*kernel[3] +
                    sdata[4*CONVCOL_W]*kernel[4];
                sdata += sStep;
            }
            gPos += gStep*4;
        }
    }
}