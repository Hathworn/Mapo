#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    int p = y * pitch + x;
    
    // Initialize shared memory to avoid branch divergence
    float val = (x < width) ? d_Data[p] : d_Data[p - x];
    minvals[tx] = val;
    maxvals[tx] = val;

    // Unroll loop for better performance
    #pragma unroll
    for (int ty = 1; ty < 16; ty++) {
        p += pitch;
        if (x < width) {
            val = d_Data[p];
            minvals[tx] = min(minvals[tx], val);
            maxvals[tx] = max(maxvals[tx], val);
        }
    }
    __syncthreads();

    // Use warp reduction to minimize synchronization
    for (int d = b / 2; d > 0; d >>= 1) {
        if (tx < d) {
            minvals[tx] = min(minvals[tx], minvals[tx + d]);
            maxvals[tx] = max(maxvals[tx], maxvals[tx + d]);
        }
        __syncthreads();
    }

    // Only one thread writes the result
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr + 0] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}