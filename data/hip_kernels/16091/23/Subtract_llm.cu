#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Calculate x and y using standard indexing.
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate linear index.
    int p = y * width + x;

    // Perform subtraction only if within bounds.
    if (x < width && y < height)
    {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}