#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS 32
#define SHARED_MEM_DATA_SIZE 16*15
#define SHARED_MEM_HIST_SIZE 32*13
#define GAUSS_SIZE 16
#define BLOCK_DIM 32

__global__ void ComputeOrientations(float *g_Data, float *d_Sift, int maxPts, int w, int h)
{
    __shared__ float data[SHARED_MEM_DATA_SIZE];
    __shared__ float hist[SHARED_MEM_HIST_SIZE];
    __shared__ float gauss[GAUSS_SIZE];
    
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    
    // Initialize histogram
    for (int i = 0; i < 13; i++) {
        hist[i * THREADS + tx] = 0.0f;
    }
    __syncthreads();
    
    // Precompute Gaussian values
    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15) {
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));
    }
    
    int xp = (int)(d_Sift[bx] - 6.5f);
    int yp = (int)(d_Sift[bx + maxPts] - 6.5f);
    int px = xp & 15;
    int x = tx - px;

    // Load data into shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = xp + x;
        int yi = yp + y;
        xi = max(0, min(xi, w - 1));
        yi = max(0, min(yi, h - 1));

        if (x >= 0 && x < 15) {
            data[memPos] = g_Data[yi * w + xi];
        }
    }
    __syncthreads();
    
    // Calculate gradients and build histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = 16.0f * atan2f(dy, dx) / 3.1416f + 16.5f;
            bin = bin == 32 ? 0 : bin;
            float grad = sqrtf(dx * dx + dy * dy);
            hist[THREADS * (x - 1) + bin] += grad * gauss[x] * gauss[y];
        }
    }
    __syncthreads();
    
    // Consolidate histogram entries
    for (int y = 0; y < 5; y++) {
        hist[y * THREADS + tx] += hist[(y + 8) * THREADS + tx];
    }
    __syncthreads();
    for (int y = 0; y < 4; y++) {
        hist[y * THREADS + tx] += hist[(y + 4) * THREADS + tx];
    }
    __syncthreads();
    for (int y = 0; y < 2; y++) {
        hist[y * THREADS + tx] += hist[(y + 2) * THREADS + tx];
    }
    __syncthreads();
    hist[tx] += hist[THREADS + tx];
    __syncthreads();

    // Gaussian smoothing
    if (tx < THREADS - 1) {
        hist[THREADS + tx] = 6 * hist[tx] + 4 * (hist[(tx + 1) % THREADS] + hist[(tx - 1 + THREADS) % THREADS]) +
                             (hist[(tx + 2) % THREADS] + hist[(tx - 2 + THREADS) % THREADS]);
    } else {
        hist[THREADS + tx] = 6 * hist[tx] + 4 * (hist[0] + hist[tx - 1]) + (hist[1] + hist[tx - 2]);
    }
    __syncthreads();

    // Find peaks in histogram
    float v = hist[THREADS + tx];
    hist[tx] = (v > hist[THREADS + ((tx + 1) % THREADS)] && v >= hist[THREADS + ((tx - 1 + THREADS) % THREADS)]) ? v : 0.0f;
    __syncthreads();

    if (tx == 0) {
        float maxval1 = 0.0f, maxval2 = 0.0f;
        int i1 = -1, i2 = -1;
        for (int i = 0; i < THREADS; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }
        
        // Subpixel precision
        float val1 = hist[THREADS + ((i1 + 1) % THREADS)];
        float val2 = hist[THREADS + ((i1 - 1 + THREADS) % THREADS)];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Sift[bx + 5 * maxPts] = 11.25f * (peak < 0.0f ? peak + THREADS : peak);
        
        if (maxval2 < 0.8f * maxval1) i2 = -1;
        
        if (i2 >= 0) {
            val1 = hist[THREADS + ((i2 + 1) % THREADS)];
            val2 = hist[THREADS + ((i2 - 1 + THREADS) % THREADS)];
            peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Sift[bx + 6 * maxPts] = 11.25f * (peak < 0.0f ? peak + THREADS : peak);
        } else {
            d_Sift[bx + 6 * maxPts] = i2;
        }
    }
}