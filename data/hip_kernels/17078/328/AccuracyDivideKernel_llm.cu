#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use atomic operation to ensure accuracy update across multiple threads.
    atomicExch(accuracy, *accuracy / N);
}