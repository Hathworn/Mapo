#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv( int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii/d3;
    z = ii % d2;
    ii = ii/d2;
    y = ii % d1;
    ii = ii/d1;
    x = ii;
    w = w*scale_factor+off_x;
    z = z*scale_factor+off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x*d1+y)*d2)+z)*d3+w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii/d3;
    z = ii % d2;
    ii = ii/d2;
    y = ii % d1;
    ii = ii/d1;
    x = ii;
    w = w/scale_factor;
    z = z/scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x*d1+y)*d2)+z)*d3+w;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x;
    ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
    if (ii >= no_elements) return;
    
    // Use iterators with single loop for better optimization
    int ipidx_base = translate_idx_inv(ii, d1, d2, d3, scale_factor, 0, 0);
    for (int i = 0, offset_x = 0, offset_y = 0; i < scale_factor * scale_factor; i++, offset_y++, offset_x = (offset_x + 1) % scale_factor) {
        int ipidx = ipidx_base + offset_x + offset_y * scale_factor * d3;
        gradInput_data[ii] += gradOutput_data[ipidx];
    }
}