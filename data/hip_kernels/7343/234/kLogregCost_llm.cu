#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        // Use fast math function for logarithm calculation
        labelLogProbs[tx] = __logf(labelp);

        // Initialize numMax and probabilistic correct class calculation directly
        if (labelp != maxp) {
            correctProbs[tx] = 0.0f;
        } else {
            int numMax = 0;
            #pragma unroll  // Unroll the loop for improved performance
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp);
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}