#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0;  // Use a local variable to accumulate results

    while (eidx < numElements) {  // Adjust loop to process all elements
        sum += a[eidx] * b[eidx];
        eidx += gridDim.x * DP_BLOCKSIZE;
    }
    shmem[threadIdx.x] = sum;  // Store local result in shared memory

    __syncthreads();

    // Use a warp-level reduction for improved efficiency
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;

        if (DP_BLOCKSIZE >= 64) mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        if (DP_BLOCKSIZE >= 32) mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        if (DP_BLOCKSIZE >= 16) mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        if (DP_BLOCKSIZE >= 8) mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        if (DP_BLOCKSIZE >= 4) mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        if (DP_BLOCKSIZE >= 2) mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    // Write the result of each block to target array
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}