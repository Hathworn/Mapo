#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use block and thread indices for parallelism, reduce operations in global memory
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index == 0) {
        *accuracy /= N;
    }
}