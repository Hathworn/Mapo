#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    // Pre-compute offsets once
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0;

    // Efficient reduction - avoid bank conflicts
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Single block reduction - reduce computation overhead
    if (idx == 0) {
        for (int i = 1; i < blockDim.x; ++i) {
            reduction_buffer[0] += reduction_buffer[i];
        }
    }
    __syncthreads();

    tmp = reduction_buffer[0];
    // Parallelize gradient calculation
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}