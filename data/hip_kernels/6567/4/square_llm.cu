#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(int *array, int arrayCount)
{
    // Use shared memory to increase cache efficiency
    extern __shared__ int sharedArray[];
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int localIdx = threadIdx.x;
    
    if (idx < arrayCount) {
        // Load from global to shared memory
        sharedArray[localIdx] = array[idx];
        __syncthreads();
        
        // Perform computation in shared memory
        sharedArray[localIdx] *= sharedArray[localIdx];
        __syncthreads();
        
        // Store results back to global memory
        array[idx] = sharedArray[localIdx];
    }
}