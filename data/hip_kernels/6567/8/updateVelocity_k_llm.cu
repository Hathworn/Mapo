#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateVelocity_k(float2 *v, float *vx, float *vy, int dx, int pdx, int dy, int lb, size_t pitch) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    float vxterm, vyterm;
    float2 nvterm;
    
    // Precalculate scale factor outside loop to optimize performance
    float scale = 1.f / (dx * dy);

    // Only proceed if within domain bounds in X
    if (gtidx < dx) {
#pragma unroll
        for (p = 0; p < lb; p++) {
            // Calculate global index for Y
            int fi = gtidy + p;
            if (fi < dy) {
                // Compute absolute global memory index
                int fjr = fi * pdx + gtidx;

                // Fetch vx and vy terms
                vxterm = vx[fjr];
                vyterm = vy[fjr];

                // Scale values
                nvterm.x = vxterm * scale;
                nvterm.y = vyterm * scale;

                // Calculate pointer to appropriate position in float2 array
                float2 *fj = (float2 *)((char *)v + fi * pitch) + gtidx;

                // Store results
                *fj = nvterm;
            }
        }
    }
}