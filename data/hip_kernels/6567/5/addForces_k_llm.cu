#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addForces_k(float2 *v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float2 *fj = (float2*)((char*)v + (ty + spy) * pitch) + tx + spx;

    float2 vterm = *fj;
    int offsetX = tx - r;  // Compute offsetX once
    int offsetY = ty - r;  // Compute offsetY once
    float offsetX4 = offsetX * offsetX * offsetX * offsetX; // Precompute offsetX^4
    float offsetY4 = offsetY * offsetY * offsetY * offsetY; // Precompute offsetY^4
    float s = 1.f / (1.f + offsetX4 + offsetY4); // Use precomputed values

    vterm.x += s * fx;
    vterm.y += s * fy;
    *fj = vterm;
}