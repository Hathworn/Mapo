#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void advectParticles_OGL(float2 *part, float2 *v, int dx, int dy, float dt, int lb, size_t pitch) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    
    // Calculate full thread and block width for loop unrolling and vectorized access
    int stride_x = blockDim.x * gridDim.x;
    int stride_y = lb * blockDim.y * gridDim.y;

    for (int x = gtidx; x < dx; x += stride_x) {
        for (int y = gtidy; y < dy; y += stride_y) {
            int fj = y * dx + x;
            float2 pterm = part[fj];

            int xvi = ((int)(pterm.x * dx));
            int yvi = ((int)(pterm.y * dy));
            float2 vterm = *((float2*)((char*)v + yvi * pitch) + xvi);

            pterm.x = fmodf(pterm.x + dt * vterm.x + 1.0f, 1.0f);  // Normalize and wrap
            pterm.y = fmodf(pterm.y + dt * vterm.y + 1.0f, 1.0f);  // Normalize and wrap

            part[fj] = pterm;
        }
    }
}