#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Apply bounds check only once
    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}