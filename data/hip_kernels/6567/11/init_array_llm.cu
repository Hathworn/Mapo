#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
    // Cache factor value into a register to avoid repetitive global memory access
    int factor_value = *factor;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = 0; i<num_iterations; i++)
        g_data[idx] += factor_value; // Access factor from register instead of global memory
}