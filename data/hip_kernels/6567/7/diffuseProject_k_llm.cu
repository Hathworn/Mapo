#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diffuseProject_k(float2 *vx, float2 *vy, int dx, int dy, float dt, float visc, int lb) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy_start = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;

    // Unrolling the loop for improved performance
    #pragma unroll
    for (int p = 0; p < lb; p++) {
        int fi = gtidy_start + p;
        
        // Use shared memory to cache data for coalesced memory access
        __shared__ float2 s_vx[32];  // Adjust size according to block dimensions
        __shared__ float2 s_vy[32];

        if (gtidx < dx && fi < dy) {
            int fj = fi * dx + gtidx;
            
            // Load data into shared memory
            s_vx[threadIdx.x] = vx[fj];
            s_vy[threadIdx.x] = vy[fj];

            float2 xterm = s_vx[threadIdx.x];
            float2 yterm = s_vy[threadIdx.x];

            int iix = gtidx;
            int iiy = (fi > dy / 2) ? (fi - dy) : fi;

            float kk = (float)(iix * iix + iiy * iiy); // k^2
            float diff = 1.f / (1.f + visc * dt * kk);
            xterm.x *= diff; xterm.y *= diff;
            yterm.x *= diff; yterm.y *= diff;

            if (kk > 0.f) {
                float rkk = 1.f / kk;
                float rkp = (iix * xterm.x + iiy * yterm.x);
                float ikp = (iix * xterm.y + iiy * yterm.y);
                xterm.x -= rkk * rkp * iix;
                xterm.y -= rkk * ikp * iix;
                yterm.x -= rkk * rkp * iiy;
                yterm.y -= rkk * ikp * iiy;
            }

            // Write back the results
            vx[fj] = xterm;
            vy[fj] = yterm;
        }
    }
}