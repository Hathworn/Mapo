#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread ID
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (idx < numEls) {
        dest[idx] = a[idx] + b[idx];
    }
}