#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//#define __OUTPUT_PIX__

#define BLOCK_SIZE 32
__constant__ __device__ float lTable_const[1064];
__constant__ __device__ float mr_const[3];
__constant__ __device__ float mg_const[3];
__constant__ __device__ float mb_const[3];

__global__ void int2lin_resmpl_messy_gpu_kernel(float *dev_in_img, float *dev_out_img, float *dev_C0_tmp, float *dev_C1_tmp, float *dev_C2_tmp, int org_wd, int org_ht, int dst_wd, int dst_ht, int n_channels, float r, int hn, int wn, int xbd0, int xbd1, int ybd0, int ybd1, int *xas_const, int *xbs_const, float *xwts_const, int *yas_const, int *ybs_const, float *ywts_const)
{
    unsigned int x_pos = threadIdx.x + (blockDim.x * blockIdx.x);
    unsigned int y_pos = threadIdx.y + (blockDim.y * blockIdx.y);

    if ((x_pos < dst_wd) && (y_pos < dst_ht)) {

        int xa, ya, yb;
        float wt, wt1;
        float *A00, *A01, *A02, *A03, *B00;
        float *A10, *A11, *A12, *A13, *B10;
        float *A20, *A21, *A22, *A23, *B20;

        float *A0 = dev_in_img + 0;
        float *B0 = dev_out_img + (0 * dst_ht * dst_wd);
        float *A1 = dev_in_img + 1;
        float *B1 = dev_out_img + (1 * dst_ht * dst_wd);
        float *A2 = dev_in_img + 2;
        float *B2 = dev_out_img + (2 * dst_ht * dst_wd);
        int y1 = 0;

        if (org_ht > dst_ht) {
            int m = 1;
            for (int iter = 0; iter < y_pos; iter++) {
                while (y1 + m < hn && yb == ybs_const[y1 + m])
                    m++;
                y1 += m;
            }
            wt = ywts_const[y1];
            wt1 = 1 - wt;
        } else {
            y1 = y_pos;
            wt = ywts_const[y1];
            wt1 = 1 - wt;
        }

        if (y_pos == 0)
            y1 = 0;

        ya = yas_const[y1];
        A00 = A0 + (ya * org_wd * n_channels);
        A01 = A00 + (org_wd * n_channels);
        A02 = A01 + (org_wd * n_channels);
        A03 = A02 + (org_wd * n_channels);

        A10 = A1 + (ya * org_wd * n_channels);
        A11 = A10 + (org_wd * n_channels);
        A12 = A11 + (org_wd * n_channels);
        A13 = A12 + (org_wd * n_channels);

        A20 = A2 + (ya * org_wd * n_channels);
        A21 = A20 + (org_wd * n_channels);
        A22 = A21 + (org_wd * n_channels);
        A23 = A22 + (org_wd * n_channels);

        yb = ybs_const[y1];
        B00 = B0 + (yb * dst_wd);
        B10 = B1 + (yb * dst_wd);
        B20 = B2 + (yb * dst_wd);

        if (x_pos < org_wd) {
            // Resample along y-direction
            if (org_ht > dst_ht) {
                int m = 1;
                while ((y1 + m < hn) && (yb == ybs_const[y1 + m]))
                    m++;

                float accum_C0 = 0.0f;
                float accum_C1 = 0.0f;
                float accum_C2 = 0.0f;
                float weight;
                for (int offset = 0; offset < m; ++offset) {
                    wt1 = ywts_const[y1 + offset];
                    accum_C0 += A00[offset * org_wd * n_channels + x_pos * n_channels] * wt1;
                    accum_C1 += A10[offset * org_wd * n_channels + x_pos * n_channels] * wt1;
                    accum_C2 += A20[offset * org_wd * n_channels + x_pos * n_channels] * wt1;
                }
                dev_C0_tmp[x_pos] = accum_C0;
                dev_C1_tmp[x_pos] = accum_C1;
                dev_C2_tmp[x_pos] = accum_C2;

            } else {
                bool yBd = y_pos < ybd0 || y_pos >= dst_ht - ybd1;
                if (yBd) {
                    dev_C0_tmp[x_pos] = A00[x_pos * n_channels];
                    dev_C1_tmp[x_pos] = A10[x_pos * n_channels];
                    dev_C2_tmp[x_pos] = A20[x_pos * n_channels];
                } else {
                    dev_C0_tmp[x_pos] = (A00[x_pos * n_channels] * wt) + (A01[x_pos * n_channels] * wt1);
                    dev_C1_tmp[x_pos] = (A10[x_pos * n_channels] * wt) + (A11[x_pos * n_channels] * wt1);
                    dev_C2_tmp[x_pos] = (A20[x_pos * n_channels] * wt) + (A21[x_pos * n_channels] * wt1);
                }
            }
        }

        __syncthreads(); // Ensure all threads have calculated C values

        if (x_pos < dst_wd) {
            // Resample along x-direction (B -> C)
            if (org_wd > dst_wd) {
                xa = xas_const[x_pos * 4];
                float accum_B0 = 0.0f;
                float accum_B1 = 0.0f;
                float accum_B2 = 0.0f;
                for (int k = 0; k < xbd0; ++k) {
                    float wt = xwts_const[(4 * x_pos) + k];
                    accum_B0 += dev_C0_tmp[xa + k] * wt;
                    accum_B1 += dev_C1_tmp[xa + k] * wt;
                    accum_B2 += dev_C2_tmp[xa + k] * wt;
                }
                B00[x_pos] = accum_B0;
                B10[x_pos] = accum_B1;
                B20[x_pos] = accum_B2;
            } else {
                int x = 0;
                for (x = 0; x < xbd0; x++) {
                    B00[x] = dev_C0_tmp[xas_const[x]] * xwts_const[x];
                    B10[x] = dev_C1_tmp[xas_const[x]] * xwts_const[x];
                    B20[x] = dev_C2_tmp[xas_const[x]] * xwts_const[x];
                }
                for (; x < dst_wd - xbd1; x++) {
                    B00[x] = dev_C0_tmp[xas_const[x]] * xwts_const[x] + dev_C0_tmp[xas_const[x] + 1] * (r - xwts_const[x]);
                    B10[x] = dev_C1_tmp[xas_const[x]] * xwts_const[x] + dev_C1_tmp[xas_const[x] + 1] * (r - xwts_const[x]);
                    B20[x] = dev_C2_tmp[xas_const[x]] * xwts_const[x] + dev_C2_tmp[xas_const[x] + 1] * (r - xwts_const[x]);
                }
                for (; x < dst_wd; x++) {
                    B00[x] = dev_C0_tmp[xas_const[x]] * xwts_const[x];
                    B10[x] = dev_C1_tmp[xas_const[x]] * xwts_const[x];
                    B20[x] = dev_C2_tmp[xas_const[x]] * xwts_const[x];
                }
            }
        }

        __syncthreads(); // Ensure all threads have completed processing
    }
}