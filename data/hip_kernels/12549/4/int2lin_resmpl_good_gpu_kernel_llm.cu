#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//#define __OUTPUT_PIX__

#define BLOCK_SIZE 32
__constant__ __device__ float lTable_const[1064];
__constant__ __device__ float mr_const[3];
__constant__ __device__ float mg_const[3];
__constant__ __device__ float mb_const[3];

__global__ void int2lin_resmpl_good_gpu_kernel(float *dev_in_img, float *dev_out_img, float *dev_C0_tmp, float *dev_C1_tmp, float *dev_C2_tmp, int org_wd, int org_ht, int dst_wd, int dst_ht, int n_channels, float r, int *yas_const, int *ybs_const)
{
    unsigned int x_pos = threadIdx.x + (blockDim.x * blockIdx.x);
    unsigned int y_pos = threadIdx.y + (blockDim.y * blockIdx.y);

    if ((x_pos < dst_wd) && (y_pos < dst_ht)) {
        int ya, yb;
        float *A00, *A01, *A02, *A03;
        float *A10, *A11, *A12, *A13;
        float *A20, *A21, *A22, *A23;
        float *B00, *B10, *B20;

        float *A0 = dev_in_img + 0;
        float *B0 = dev_out_img + (0 * dst_ht * dst_wd);
        float *A1 = dev_in_img + 1;
        float *B1 = dev_out_img + (1 * dst_ht * dst_wd);
        float *A2 = dev_in_img + 2;
        float *B2 = dev_out_img + (2 * dst_ht * dst_wd);

        if (org_ht == dst_ht && org_wd == dst_wd) {
            int out_img_idx = y_pos + (dst_wd * x_pos);
            B0[out_img_idx] = A0[out_img_idx * n_channels];
            B1[out_img_idx] = A1[out_img_idx * n_channels];
            B2[out_img_idx] = A2[out_img_idx * n_channels];
            return;
        }

        int y1 = 0;
        y1 += (org_ht / dst_ht) * y_pos;

        if (y_pos == 0) y1 = 0;

        ya = yas_const[y1];
        A00 = A0 + (ya * org_wd * n_channels);
        A01 = A00 + (org_wd * n_channels);
        A02 = A01 + (org_wd * n_channels);
        A03 = A02 + (org_wd * n_channels);

        A10 = A1 + (ya * org_wd * n_channels);
        A11 = A10 + (org_wd * n_channels);
        A12 = A11 + (org_wd * n_channels);
        A13 = A12 + (org_wd * n_channels);

        A20 = A2 + (ya * org_wd * n_channels);
        A21 = A20 + (org_wd * n_channels);
        A22 = A21 + (org_wd * n_channels);
        A23 = A22 + (org_wd * n_channels);

        yb = ybs_const[y1];
        B00 = B0 + (yb * dst_wd);
        B10 = B1 + (yb * dst_wd);
        B20 = B2 + (yb * dst_wd);

        // Resample along y direction
        int ht_ratio = org_ht / dst_ht;
        dev_C0_tmp[x_pos] = 0;
        dev_C1_tmp[x_pos] = 0;
        dev_C2_tmp[x_pos] = 0;
        for (int i = 0; i < ht_ratio; i++) {
            dev_C0_tmp[x_pos] += (A00 + i * (org_wd * n_channels))[x_pos * n_channels];
            dev_C1_tmp[x_pos] += (A10 + i * (org_wd * n_channels))[x_pos * n_channels];
            dev_C2_tmp[x_pos] += (A20 + i * (org_wd * n_channels))[x_pos * n_channels];
        }

        __syncthreads();

        // Resample along x direction (B -> C)
        int wd_ratio = org_wd / dst_wd;
        B00[x_pos] = 0;
        B10[x_pos] = 0;
        B20[x_pos] = 0;
        for (int i = 0; i < wd_ratio; i++) {
            B00[x_pos] += dev_C0_tmp[wd_ratio * x_pos + i];
            B10[x_pos] += dev_C1_tmp[wd_ratio * x_pos + i];
            B20[x_pos] += dev_C2_tmp[wd_ratio * x_pos + i];
        }

        float scale = r / (ht_ratio * wd_ratio);
        B00[x_pos] *= scale;
        B10[x_pos] *= scale;
        B20[x_pos] *= scale;

        __syncthreads();
    }
}