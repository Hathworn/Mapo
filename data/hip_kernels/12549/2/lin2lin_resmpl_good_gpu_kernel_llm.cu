#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//#define __OUTPUT_PIX__

#define BLOCK_SIZE 32
__constant__ __device__ float lTable_const[1064];
__constant__ __device__ float mr_const[3];
__constant__ __device__ float mg_const[3];
__constant__ __device__ float mb_const[3];

__global__ void lin2lin_resmpl_good_gpu_kernel(float *dev_in_img, float *dev_out_img, float *dev_C0_tmp, float *dev_C1_tmp, float *dev_C2_tmp, int org_wd, int org_ht, int dst_wd, int dst_ht, int n_channels, float r, int *yas_const, int *ybs_const)
{
    unsigned int x_pos = threadIdx.x + (blockDim.x * blockIdx.x);
    unsigned int y_pos = threadIdx.y + (blockDim.y * blockIdx.y);

    if ((x_pos < dst_wd) && (y_pos < dst_ht)) {
        int ya, yb;
        float *A0 = dev_in_img + (0 * org_ht * org_wd);
        float *B0 = dev_out_img + (0 * dst_ht * dst_wd);
        float *A1 = dev_in_img + (1 * org_ht * org_wd);
        float *B1 = dev_out_img + (1 * dst_ht * dst_wd);
        float *A2 = dev_in_img + (2 * org_ht * org_wd);
        float *B2 = dev_out_img + (2 * dst_ht * dst_wd);

        // Direct copy if no need for resampling
        if (org_ht == dst_ht && org_wd == dst_wd) {
            int out_img_idx = y_pos * dst_wd + x_pos;
            B0[out_img_idx] = A0[out_img_idx * n_channels];
            B1[out_img_idx] = A1[out_img_idx * n_channels];
            B2[out_img_idx] = A2[out_img_idx * n_channels];
            return;
        }

        // Calculate scaled y position
        int y1 = y_pos * (org_ht / dst_ht);

        ya = yas_const[y1];
        float *A00 = A0 + (ya * org_wd);
        float *A10 = A1 + (ya * org_wd);
        float *A20 = A2 + (ya * org_wd);

        yb = ybs_const[y1];
        float *B00 = B0 + (yb * dst_wd);
        float *B10 = B1 + (yb * dst_wd);
        float *B20 = B2 + (yb * dst_wd);

        // Resample along y direction
        int y_ratio = org_ht / dst_ht;
        float c0 = 0, c1 = 0, c2 = 0;
        for (int i = 0; i < y_ratio; ++i) {
            c0 += A00[x_pos + i * org_wd];
            c1 += A10[x_pos + i * org_wd];
            c2 += A20[x_pos + i * org_wd];
        }
        dev_C0_tmp[x_pos] = c0;
        dev_C1_tmp[x_pos] = c1;
        dev_C2_tmp[x_pos] = c2;

        __syncthreads();

        // Resample along x direction (B -> C)
        int x_ratio = org_wd / dst_wd;
        float sum0 = 0, sum1 = 0, sum2 = 0;
        for (int i = 0; i < x_ratio; ++i) {
            sum0 += dev_C0_tmp[x_ratio * x_pos + i];
            sum1 += dev_C1_tmp[x_ratio * x_pos + i];
            sum2 += dev_C2_tmp[x_ratio * x_pos + i];
        }
        B00[x_pos] = sum0 * (r / x_ratio);
        B10[x_pos] = sum1 * (r / x_ratio);
        B20[x_pos] = sum2 * (r / x_ratio);
        
        __syncthreads();
    }
}