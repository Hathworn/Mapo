#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//#define __OUTPUT_PIX__

#define BLOCK_SIZE 32
__constant__ __device__ float lTable_const[1064];
__constant__ __device__ float mr_const[3];
__constant__ __device__ float mg_const[3];
__constant__ __device__ float mb_const[3];

__global__ void trianguler_convolution_gpu_kernel(float *dev_I, float *dev_O, float *T0, float *T1, float *T2, int wd, int ht, float nrm, float p) {
    unsigned int x_pos = threadIdx.x + (blockDim.x * blockIdx.x);
    unsigned int y_pos = threadIdx.y + (blockDim.y * blockIdx.y);

    if ((x_pos < wd) && (y_pos < ht)) {
        // Use registers for intermediate calculations
        float It0, It1, It2, Im0, Im1, Im2, Ib0, Ib1, Ib2;
        float Ot0, Ot1, Ot2;
        float T00, T10, T20;

        // Compute read offsets
        unsigned int offset = (y_pos * wd) + x_pos;
        unsigned int offset_prev_row = offset - wd;
        unsigned int offset_next_row = offset + wd;

        // Read input
        Im0 = dev_I[offset];
        Im1 = dev_I[offset + ht * wd];
        Im2 = dev_I[offset + 2 * ht * wd];

        if (y_pos > 0) {
            It0 = dev_I[offset_prev_row];
            It1 = dev_I[offset_prev_row + ht * wd];
            It2 = dev_I[offset_prev_row + 2 * ht * wd];
        } else {
            It0 = Im0; It1 = Im1; It2 = Im2;
        }

        if (y_pos < ht - 1) {
            Ib0 = dev_I[offset_next_row];
            Ib1 = dev_I[offset_next_row + ht * wd];
            Ib2 = dev_I[offset_next_row + 2 * ht * wd];
        } else {
            Ib0 = Im0; Ib1 = Im1; Ib2 = Im2;
        }

        // Perform convolution
        T00 = nrm * (It0 + (p * Im0) + Ib0);
        T10 = nrm * (It1 + (p * Im1) + Ib1);
        T20 = nrm * (It2 + (p * Im2) + Ib2);

        __syncthreads(); // Ensure all threads have calculated T values

        // Compute output
        if (x_pos == 0) {
            Ot0 = ((1 + p) * T00) + T0[offset + 1];
            Ot1 = ((1 + p) * T10) + T1[offset + 1];
            Ot2 = ((1 + p) * T20) + T2[offset + 1];
        } else if (x_pos == wd - 1) {
            Ot0 = T0[offset - 1] + ((1 + p) * T00);
            Ot1 = T1[offset - 1] + ((1 + p) * T10);
            Ot2 = T2[offset - 1] + ((1 + p) * T20);
        } else {
            Ot0 = T0[offset - 1] + (p * T00) + T0[offset + 1];
            Ot1 = T1[offset - 1] + (p * T10) + T1[offset + 1];
            Ot2 = T2[offset - 1] + (p * T20) + T2[offset + 1];
        }

        // Write output
        dev_O[offset] = Ot0;
        dev_O[offset + ht * wd] = Ot1;
        dev_O[offset + 2 * ht * wd] = Ot2;

        __syncthreads();
    }
}