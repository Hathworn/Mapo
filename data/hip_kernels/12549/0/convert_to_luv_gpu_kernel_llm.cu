#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//#define __OUTPUT_PIX__

#define BLOCK_SIZE 32
__constant__ __device__ float lTable_const[1064];
__constant__ __device__ float mr_const[3];
__constant__ __device__ float mg_const[3];
__constant__ __device__ float mb_const[3];

__global__ void convert_to_luv_gpu_kernel(unsigned char *in_img, float *out_img, int cols, int rows, bool use_rgb)
{
    // Calculate the pixel position in the image
    unsigned int x_pos = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y_pos = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x_pos < cols) && (y_pos < rows)) {
        unsigned int pos = (y_pos * cols) + x_pos;
        
        // Instead of multiple global memory accesses, using registers for r, g, b
        float r = (float)in_img[(3 * pos) + (use_rgb ? 0 : 2)];
        float g = (float)in_img[(3 * pos) + 1];
        float b = (float)in_img[(3 * pos) + (use_rgb ? 2 : 0)];

        // Direct transformation to target variables
        float x = (mr_const[0] * r) + (mg_const[0] * g) + (mb_const[0] * b);
        float y = (mr_const[1] * r) + (mg_const[1] * g) + (mb_const[1] * b);
        float z = (mr_const[2] * r) + (mg_const[2] * g) + (mb_const[2] * b);

        // Precompute constants outside the critical path
        const float maxi = 1.0f / 270;
        const float minu = -88.0f * maxi;
        const float minv = -134.0f * maxi;
        const float un = 0.197833f;
        const float vn = 0.468331f;

        // Luv conversion with fused multiply-add (FMA) style operations
        float lt = lTable_const[static_cast<int>((y * 1024))];
        float l = lt;
        z = 1.0f / (x + (15.0f * y) + (3.0f * z) + 1e-35f);
        float u = (lt * (13.0f * 4.0f * x * z - 13.0f * un)) - minu;
        float v = (lt * (13.0f * 9.0f * y * z - 13.0f * vn)) - minv;

        // Output the results
        out_img[(3 * pos)] = l;
        out_img[(3 * pos) + 1] = u;
        out_img[(3 * pos) + 2] = v;
    }
}