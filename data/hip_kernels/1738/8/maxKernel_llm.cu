#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void maxKernel(float *array, int size, float* max)
{
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + tid;
    int stride = blockDim.x * 2 * gridDim.x;

    // Initialize shared memory
    sdata[tid] = (i < size) ? array[i] : -FLT_MAX;
    if (i + blockDim.x < size) {
        sdata[tid] = fmaxf(sdata[tid], array[i + blockDim.x]);
    }
    __syncthreads();

    // Reduce using shared memory
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] = fmaxf(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    // Unrolling warp-level reduction
    if (tid < 32) {
        volatile float *vsmem = sdata;
        vsmem[tid] = fmaxf(vsmem[tid], vsmem[tid + 32]);
        vsmem[tid] = fmaxf(vsmem[tid], vsmem[tid + 16]);
        vsmem[tid] = fmaxf(vsmem[tid], vsmem[tid + 8]);
        vsmem[tid] = fmaxf(vsmem[tid], vsmem[tid + 4]);
        vsmem[tid] = fmaxf(vsmem[tid], vsmem[tid + 2]);
        vsmem[tid] = fmaxf(vsmem[tid], vsmem[tid + 1]);
    }

    // Write result for this block to the max array
    if (tid == 0) {
        max[blockIdx.x] = sdata[0];
    }
}