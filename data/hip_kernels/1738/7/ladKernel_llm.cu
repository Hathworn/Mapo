#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ladKernel(float *a, float *b, float *out, int size) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    int stride = blockDim.x * 2 * gridDim.x;
    sdata[tid] = 0;

    // Unrolling the loop for performance improvement
    while (i < size) {
        sdata[tid] += abs(a[i] - b[i]) + abs(a[i + blockDim.x] - b[i + blockDim.x]);
        i += stride;
    }
    __syncthreads();

    // Optimize reduction process
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Unrolling warp reduction
    if (tid < 32) {
        volatile float* smem = sdata; // Use volatile to prevent reordering
        smem[tid] += smem[tid + 32];
        smem[tid] += smem[tid + 16];
        smem[tid] += smem[tid + 8];
        smem[tid] += smem[tid + 4];
        smem[tid] += smem[tid + 2];
        smem[tid] += smem[tid + 1];
    }
    
    if (tid == 0) {
        out[blockIdx.x] = sdata[0];
    }
}