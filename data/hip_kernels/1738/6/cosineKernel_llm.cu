#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cosineKernel(float *a, float *b, float *outN, float *outD1, float *outD2, int size) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;

    float sum0 = 0.0f; // Initialize local sums
    float sum1 = 0.0f;
    float sum2 = 0.0f;

    int stride = gridDim.x * blockDim.x;
    
    while (i < size) {
        // Compute partial results for pair of elements
        float val0 = a[i] * b[i];
        float val1 = a[i + blockDim.x] * b[i + blockDim.x];

        // Accumulate to local sums
        sum0 += val0 + val1;
        sum1 += val0 + val1;
        sum2 += val0 + val1;

        i += stride;
    }

    // Store local sums into shared memory
    sdata[3 * tid]     = sum0;
    sdata[3 * tid + 1] = sum1;
    sdata[3 * tid + 2] = sum2;

    __syncthreads();

    // Perform parallel reduction within block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[3 * tid]     += sdata[3 * tid + s];
            sdata[3 * tid + 1] += sdata[3 * tid + s + 1];
            sdata[3 * tid + 2] += sdata[3 * tid + s + 2];
        }
        __syncthreads(); // Ensure all threads have updated before next reduction step
    }

    // Write result for each block to global memory
    if (tid == 0) {
        outN[blockIdx.x] = sdata[0];
        outD1[blockIdx.x] = sdata[1];
        outD2[blockIdx.x] = sdata[2];
    }
}