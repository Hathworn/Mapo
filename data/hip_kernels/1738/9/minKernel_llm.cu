#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void minKernel(float *array, int size, float* min)
{
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    int stride = blockDim.x * 2 * gridDim.x;

    // Load initial elements into shared memory; handling boundary condition
    if (i < size) {
        sdata[tid] = fminf(array[i], (i + blockDim.x < size) ? array[i + blockDim.x] : array[i]);
    } else {
        sdata[tid] = FLT_MAX; // Use FLT_MAX as a neutral minimum for unused threads
    }
    __syncthreads();

    // Reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = fminf(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        min[blockIdx.x] = sdata[0];
    }
}