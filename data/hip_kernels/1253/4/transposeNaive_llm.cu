#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaive(float *odata, float *idata, int width, int height)
{
    // Calculate global thread indices
    int xIndex = blockIdx.x * TILE_DIM + threadIdx.x;
    int yIndex = blockIdx.y * TILE_DIM + threadIdx.y;

    // Check boundary conditions
    if (xIndex < width && yIndex < height) {
        int index_in = xIndex + width * yIndex;

        // Unroll the loop for better performance
        for (int i = 0; i < TILE_DIM; i += BLOCK_ROWS) {
            int index_out = (yIndex + i) + height * xIndex;
            if (yIndex + i < height) {  // Boundary check for accesses
                odata[index_out] = idata[index_in + i * width];
            }
        }
    }
}