#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Use shared memory to optimize memory access patterns
    __shared__ float sharedA[256];
    __shared__ float sharedB[256];

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        // Load data into shared memory
        sharedA[threadIdx.x] = a[i];
        sharedB[threadIdx.x] = b[i];
        __syncthreads(); // Ensure all threads have loaded their data

        // Perform the computation
        dest[i] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
        __syncthreads(); // Ensure all threads have written their results
    }
}