#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Flattened unique thread index
    if(index >= N) return;

    int b = index / (spatial * layers); // Batch index
    int rem = index % (spatial * layers);
    int in_c = rem / spatial; // Channel index
    int in_s = rem % spatial; // Spatial index

    int i1 = b * layers * spatial + in_c * spatial + in_s; // Input index calculation
    int i2 = b * spatial * layers + in_s * layers + in_c; // Transposed index calculation

    if (forward) {
        out[i2] = x[i1]; // Write transposed data
    } else {
        out[i1] = x[i2]; // Write non-transposed data
    }
}