#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Optimize looping by iterating over flattened index space
    for (int i = index; i < N; i += stride) {
        int f = (i / spatial) % filters;
        x[i] = (x[i] - mean[f]) / (sqrt(variance[f] + .00001f));
    }
}