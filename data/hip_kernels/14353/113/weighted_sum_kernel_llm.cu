#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate global index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Process elements within bounds
    if (i < n) {
        // Use ternary operator to handle b being null directly within calculation
        c[i] = s[i] * a[i] + (1.0f - s[i]) * (b ? b[i] : 0.0f);
    }
}