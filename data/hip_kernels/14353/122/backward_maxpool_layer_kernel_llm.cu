#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes) {

    // Calculate output dimensions
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;
    int area = (size - 1) / stride;

    // Calculate unique thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Calculate original indices
    int index = id;
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0;

    // Loop over the pooling window
    for (int l = -area; l <= area; ++l) {
        for (int m = -area; m <= area; ++m) {
            int out_w = (j + w_offset) / stride + m;  // Corrected calculation
            int out_h = (i + h_offset) / stride + l;  // Corrected calculation
            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h) {
                int out_index = out_w + w * (out_h + h * (k + c * b));
                d += (indexes[out_index] == index) ? delta[out_index] : 0;
            }
        }
    }
    atomicAdd(&prev_delta[index], d);  // Use atomic operation for thread safety
}