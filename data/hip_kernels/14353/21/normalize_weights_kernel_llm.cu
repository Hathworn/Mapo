#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float_t d_randu(int * seed, int index) {
    int M = INT_MAX;
    int A = 1103515245;
    int C = 12345;
    int num = A * seed[index] + C;
    seed[index] = num % M;
    return fabsf(seed[index] / ((float_t) M));
}

__device__ void cdfCalc(float_t * CDF, float_t * weights, int Nparticles) {
    int x;
    CDF[0] = weights[0];
    for (x = 1; x < Nparticles; x++) {
        CDF[x] = weights[x] + CDF[x - 1];
    }
}

__global__ void normalize_weights_kernel(float_t * weights, int Nparticles, float_t* partial_sums, float_t * CDF, float_t * u, int * seed) {
    int block_id = blockIdx.x;
    int i = blockDim.x * block_id + threadIdx.x;
    __shared__ float_t u1, sumWeights;

    // Calculate sumWeights once per block
    if (0 == threadIdx.x)
        sumWeights = partial_sums[0];
    __syncthreads();

    // Normalize weights using precomputed sumWeights
    if (i < Nparticles) {
        weights[i] /= sumWeights;
    }
    __syncthreads();

    // Compute cumulative distribution function
    if (i == 0) {
        cdfCalc(CDF, weights, Nparticles);
        u[0] = (1 / ((float_t) (Nparticles))) * d_randu(seed, i);
    }
    __syncthreads();

    // Load u1 once per block
    if (0 == threadIdx.x)
        u1 = u[0];
    __syncthreads();

    // Compute resampled u values
    if (i < Nparticles) {
        u[i] = u1 + i / ((float_t) (Nparticles));
    }
}