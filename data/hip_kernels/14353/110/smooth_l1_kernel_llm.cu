#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;  // Calculate stride for grid-stride loop

    // Use grid-stride loop for improved parallelism
    for (int idx = i; idx < n; idx += stride) {
        float diff = truth[idx] - pred[idx];
        float abs_val = fabsf(diff);  // Use faster floating-point absolute value function
        if (abs_val < 1.0f) {
            error[idx] = diff * diff;
            delta[idx] = diff;
        } else {
            error[idx] = 2.0f * abs_val - 1.0f;
            delta[idx] = (diff > 0.0f) ? 1.0f : -1.0f;
        }
    }
}