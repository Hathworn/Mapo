#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Load X[i*INCX] to a register to avoid redundant memory access
    if (i < N) {
        float xi = X[i * INCX];
        if ((xi * xi) < (ALPHA * ALPHA)) {
            X[i * INCX] = 0;
        }
    }
}