#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relabel2Kernel(int *components, int previousLabel, int newLabel, const int colsComponents, const int idx, const int frameRows) {
    // Calculate global index using block and thread indices
    uint globalIdx = (blockIdx.x * blockDim.x + threadIdx.x) * colsComponents + 
                     (blockIdx.y * blockDim.y + threadIdx.y) + 
                     colsComponents * frameRows * idx;
                     
    // Conditional branch to update labels
    if (components[globalIdx] == previousLabel) {
        components[globalIdx] = newLabel;
    }
}