#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index with shared memory optimization
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N) {
        int idx = i * INCX;
        X[idx] *= ALPHA;
    }
}