#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    int b = id / c; // Optimized division for better performance

    int out_index = k + c * b;
    float scale = 1.0f / (w * h); // Pre-compute scale factor

    // Unrolling the loop for better performance
    int steps = w * h;
    for(int i = 0; i < steps; i++) {
        int in_index = i + w * h * out_index;
        in_delta[in_index] += out_delta[out_index] * scale;
    }
}