#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relabelKernel(int *components, int previousLabel, int newLabel, const int colsComponents) {
    // Calculate the linear index in a single statement for improved performance
    uint index = (blockIdx.x * blockDim.x + threadIdx.x) * colsComponents + (blockIdx.y * blockDim.y + threadIdx.y;
    
    // Use inline conditional check to avoid branch divergence
    if (components[index] == previousLabel) {
        components[index] = newLabel;
    }
}