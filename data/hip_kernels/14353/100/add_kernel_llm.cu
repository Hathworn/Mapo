#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX) {
    // Calculate global thread index using grid-stride loop approach
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) {
        X[i * INCX] += ALPHA;  // Perform the operation only if within bounds
    }
}