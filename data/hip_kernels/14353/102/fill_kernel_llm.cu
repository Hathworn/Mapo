#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Compute global thread ID
    int grid_stride = gridDim.x * blockDim.x;       // Calculate grid stride

    for (; i < N; i += grid_stride) {               // Iterate over all elements processed by this thread
        X[i * INCX] = ALPHA;                        // Set the value
    }
}