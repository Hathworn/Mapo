#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    // Calculate global index
    int s = blockIdx.x * blockDim.x + threadIdx.x;

    // Return if index exceeds size
    if (s >= size) return;

    // Utilize shared memory for mean calculation to reduce global memory access
    extern __shared__ float shared_input[];
    float mean = 0.0f;

    // Store input data in shared memory
    for (int i = 0; i < n; ++i) {
        shared_input[i] = abs(input[i * size + s]);
    }

    // Compute mean using shared memory
    for (int i = 0; i < n; ++i) {
        mean += shared_input[i];
    }
    mean /= n;

    // Assign to binary array
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}