#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D grid for simplicity
    int stride = gridDim.x * blockDim.x; // Calculate stride for loop unrolling
    while (i < N) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX]; // Perform computation
        i += stride; // Move to next data chunk
    }
}