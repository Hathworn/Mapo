#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Use shared memory for faster access
    extern __shared__ float shared_mean[];

    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Load mean into shared memory
    if (threadIdx.x < filters) {
        shared_mean[threadIdx.x] = mean[threadIdx.x];
    }
    __syncthreads();  

    float scale = 1.0f / (batch * spatial - 1);
    float var_sum = 0.0f;

    // Reduce repeated reads by storing mean in shared memory
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            var_sum += pow((x[index] - shared_mean[i]), 2);
        }
    }

    variance[i] = var_sum * scale;
}