#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Calculate unique global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure that the thread is within bounds
    if (i < groups) {
        float group_sum = 0.0f;  // Use a register to hold the accumulating sum
        for (int k = 0; k < n; ++k) { 
            group_sum += x[k * groups + i];
        }
        sum[i] = group_sum;  // Write the computed sum to the output array
    }
}