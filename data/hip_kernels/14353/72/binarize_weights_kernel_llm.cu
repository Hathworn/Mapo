#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure valid range for f
    if (f >= n) return;
    
    // Initializes the mean variable
    float mean = 0;
    
    // Use shared memory to reduce global memory access latency
    extern __shared__ float shared_weights[];
    
    // Load weights into shared memory
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[i] = abs(weights[f * size + i]);
    }
    
    // Synchronize threads before using shared memory results
    __syncthreads();
    
    // Accumulate mean in shared memory to optimize access
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        mean += shared_weights[i];
    }
    
    // Reduce total mean using a block-wide reduction and synchronize
    atomicAdd(&mean, shared_weights[threadIdx.x]);
    __syncthreads();
    
    // Calculate mean once per thread
    mean = mean / size;
    
    // Binarization using calculated mean
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }
}