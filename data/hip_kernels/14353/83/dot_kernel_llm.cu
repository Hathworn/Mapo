#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  // Stride for grid-striding loop

    for (; index < batch * n * (n - 1) / 2 * size; index += stride) {
        int f1 = (int)floor((-1 + sqrt(1 + 8 * index / size)) / 2);
        int f2 = index / size - (f1 * (f1 + 1)) / 2;
        if (f2 <= f1) continue;

        float sum = 0.0f;
        float norm1 = 0.0f;
        float norm2 = 0.0f;

        // Unroll loops for better performance if possible
        for (int b = 0; b < batch; ++b) {
            for (int i = 0; i < size; ++i) {
                int i1 = b * size * n + f1 * size + i;
                int i2 = b * size * n + f2 * size + i;
                float val1 = output[i1];
                float val2 = output[i2];

                sum += val1 * val2;
                norm1 += val1 * val1;
                norm2 += val2 * val2;
            }
        }

        norm1 = rsqrtf(norm1);  // Use rsqrtf for faster calculation
        norm2 = rsqrtf(norm2);
        float norm = norm1 * norm2 / sum;
        
        for (int b = 0; b < batch; ++b) {
            for (int i = 0; i < size; ++i) {
                int i1 = b * size * n + f1 * size + i;
                int i2 = b * size * n + f2 * size + i;
                float val = scale * norm;
                atomicAdd(&delta[i1], -val * output[i2]);
                atomicAdd(&delta[i2], -val * output[i1]);
            }
        }
    }
}