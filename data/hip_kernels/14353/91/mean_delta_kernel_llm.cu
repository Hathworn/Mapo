#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Use local accumulator to minimize global memory access
    float local_sum = 0;

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            local_sum += delta[index];
        }
    }

    // Store the result of local sum after scaling
    mean_delta[i] = local_sum * (-1.f / sqrtf(variance[i] + 0.00001f));
}