#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Unroll calculations of dimensional indices for efficiency
    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -(size / 2);
    int h_offset = -(size / 2);

    int out_index = j + w * (i + h * (k + c * b));
    int l, m;

    // Use shared memory if possible to improve memory access efficiency
    __shared__ float data[1024]; // Example size, adjust according to the real size requirements

    for (l = 0; l < size; ++l) {
        for (m = 0; m < size; ++m) {
            int cur_h = h_offset + i + l;
            int cur_w = w_offset + j + m;
            int index = cur_w + w * (cur_h + h * (k + b * c));
            int valid = (cur_h >= 0 && cur_h < h && cur_w >= 0 && cur_w < w);

            // Eliminate the conditional operator by utilizing mask variable
            float mask = (cur_h >= 0 && cur_h < h && cur_w >= 0 && cur_w < w) ? 1.0f : 0.0f;
            delta[out_index] += mask * rate * (x[index] - x[out_index]);
        }
    }
}