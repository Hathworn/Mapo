#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    int f = (index / spatial) % filters;

    // Pre-compute reusable values
    float variance_sqrt_inv = rsqrtf(variance[f] + .00001f);
    float scale = variance_delta[f] * 2.0f / (spatial * batch);

    // Update delta
    delta[index] = delta[index] * variance_sqrt_inv + scale * (x[index] - mean[f]) + mean_delta[f] / (spatial * batch);
}