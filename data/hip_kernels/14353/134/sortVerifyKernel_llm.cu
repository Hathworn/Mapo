#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sortVerifyKernel(uint *d_DstKey, uint *d_DstVal, uint *d_SrcKey, uint *errNum)
{
    // Calculate global index
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory to reduce global memory access
    __shared__ uint sharedSrcKey[THREAD_SIZE];
    __shared__ uint sharedDstVal[THREAD_SIZE];

    // Load data from global to shared memory
    for (uint iterator = threadIdx.x; iterator < THREAD_SIZE; iterator += blockDim.x) {
        sharedDstVal[iterator] = d_DstVal[idx * THREAD_SIZE + iterator];
        sharedSrcKey[iterator] = d_SrcKey[sharedDstVal[iterator]];
    }
    __syncthreads();

    // Verify sorted key
    for (uint iterator = 0; iterator < THREAD_SIZE; iterator++) {
        if (sharedSrcKey[iterator] != d_DstKey[idx * THREAD_SIZE + iterator]) {
            atomicAdd(errNum, 1);
        }
    }
}