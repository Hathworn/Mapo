#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    
    // Calculate variance scaling factor once, outside the loop
    float var_scale = -.5 * pow(variance[i] + .00001f, (float)(-3./2.));
    
    float sum = 0; // Use a local variable for accumulation
    for(int j = 0; j < batch; ++j){
#pragma unroll  // Potentially improve loop performance
        for(int k = 0; k < spatial; ++k){
            int index = j * filters * spatial + i * spatial + k;
            sum += delta[index] * (x[index] - mean[i]);
        }
    }
    variance_delta[i] = sum * var_scale; // Apply computed scaling
}