#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate the unique thread index within the grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index is within bounds
    if (i < N) {
        // Perform multiplication
        Y[i * INCY] *= X[i * INCX];
    }
}