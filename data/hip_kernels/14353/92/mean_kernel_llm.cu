#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Calculate scale multiplier once
    float scale = 1.0f / (batch * spatial);
    
    // Compute global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) {
        return; // Early exit if the index is out of bounds
    }

    mean[i] = 0.0f;

    // Unroll the inner loop to improve memory access patterns
    for (int j = 0; j < batch; ++j) {
        int base_index = j * filters * spatial + i * spatial;
        for (int k = 0; k < spatial; ++k) {
            mean[i] += x[base_index + k];
        }
    }

    mean[i] *= scale; // Apply scaling only once
}