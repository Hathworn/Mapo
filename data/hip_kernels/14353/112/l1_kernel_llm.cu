#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index with one less multiplication
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a stride loop to manage excess elements beyond the grid size
    while(i < n) {
        float diff = truth[i] - pred[i];
        // Use fabsf for floating-point absolute value
        error[i] = fabsf(diff);
        // Use ternary operator directly
        delta[i] = (diff > 0) ? 1.0f : -1.0f;
        
        // Increment index by total number of threads
        i += blockDim.x * gridDim.x;
    }
}