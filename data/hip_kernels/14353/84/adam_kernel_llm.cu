#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    // Calculate unique thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Return if index is out of bounds
    if (index >= N) return;
    
    // Precompute powers and square root for efficiency
    float B2_t = pow(B2, t);
    float B1_t = pow(B1, t);
    float sqrt_v = sqrt(v[index]);
    
    // Update x using precomputed values
    x[index] += rate * sqrt(1.0f - B2_t) / (1.0f - B1_t) * m[index] / (sqrt_v + eps);
}