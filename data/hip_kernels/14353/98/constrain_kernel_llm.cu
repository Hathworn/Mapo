#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index for each thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Stride through the array if the total number of elements is more than the total number of threads
    for (; i < N; i += blockDim.x * gridDim.x)
    {
        // Constrain each element to the range [-ALPHA, ALPHA]
        X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i * INCX]));
    }
}