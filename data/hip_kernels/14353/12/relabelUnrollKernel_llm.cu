#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relabelUnrollKernel(int *components, int previousLabel, int newLabel, const int colsComponents, const int idx, const int frameRows, const int factor) {
    // Use the linear thread id as vectorized index calculation
    uint id_i_child = (blockIdx.x * blockDim.x) + threadIdx.x + (frameRows * idx);
    uint id_j_child = blockIdx.y * blockDim.y + threadIdx.y;
    uint stride = gridDim.y * blockDim.y; // Calculate stride for coalesced access

    // Iterate with stride for better memory coalescing
    for (int j = id_j_child; j < (colsComponents / factor); j += stride) {
        int index = i * colsComponents + j; // Computation done once
        if (components[index] == previousLabel) {
            components[index] = newLabel;
        }
    }
}