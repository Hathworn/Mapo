#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = gridDim.x * blockDim.x;
    
    // Use grid-stride loop to optimize accesses and utilize more threads
    for (int idx = i; idx < N; idx += gridStride) {
        Y[idx * INCY] = powf(X[idx * INCX], ALPHA);
    }
}