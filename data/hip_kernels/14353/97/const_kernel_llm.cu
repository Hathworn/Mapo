#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Optimize with loop unrolling for coalesced memory access
    int stride = blockDim.x * gridDim.x * INCX;
    for (int idx = i * INCX; idx < N; idx += stride) {
        X[idx] = ALPHA;
    }
}