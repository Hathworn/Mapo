#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate global thread index
    int i = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x;

    // Ensure valid thread index within bounds
    if (i < n) {
        // Initialize pointers to access memory coalesced and avoid branch divergence
        float dc_val = dc[i];
        float s_val = s[i];
        float a_val = a[i];
        float b_val = b[i];
        
        // Use direct index with cached values for better memory access
        if (da) da[i] += dc_val * s_val;
        db[i] += dc_val * (1 - s_val);
        ds[i] += dc_val * (a_val - b_val);
    }
}