#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate only once to avoid redundant calculations
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    // Ensure thread handles valid index within bounds
    if(index < n)
    {
        float diff = truth[index] - pred[index];
        
        // Compute error and delta in a streamlined manner
        error[index] = diff * diff;
        delta[index] = diff;
    }
}