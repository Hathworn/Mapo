#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Early return if index is out of bounds
    if (i >= N) return;

    // Precompute common divisions to reduce redundant calculations
    int wh = w * h;
    int wc = w * c;
    int hc = h * c;
    int stride2 = stride * stride;
    int out_c = c / stride2;

    // Decompose index into b, c, h, w
    int in_index = i;
    int b = i / wc;
    i %= wc;
    int in_c = i / wh;
    i %= wh;
    int in_h = i / w;
    int in_w = i % w;

    // Offset calculations for output
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;

    // Calculate output index
    int out_index = (b * out_c + c2) * stride * stride * wh + h2 * stride * w + w2;

    // Perform operation based on 'forward' flag
    if (forward) {
        out[out_index] = x[in_index];
    } else {
        out[in_index] = x[out_index];
    }
}