#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate unique thread index within the grid
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    // Ensure boundary checks to avoid accessing out-of-bound memory
    if(i < N) {
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}