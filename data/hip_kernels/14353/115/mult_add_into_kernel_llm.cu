#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Flatten 1D block
    int stride = blockDim.x * gridDim.x;           // Calculate stride

    // Loop to process multiple elements per thread, improving memory coalescing
    for (; i < n; i += stride) {
        c[i] += a[i] * b[i];
    }
}