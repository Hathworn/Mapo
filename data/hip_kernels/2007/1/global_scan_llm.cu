#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void global_scan(float* d_out, float* d_in, int num_elements)
{
    int index = threadIdx.x; // Get thread index
    int stride;
    float temp;

    if (index < num_elements) // Ensure threads do not access out-of-bound memory
    {
        d_out[index] = d_in[index];
    }
    __syncthreads();

    for (stride = 1; stride < num_elements; stride *= 2) // Iterate through power of 2 strides
    {
        if (index >= stride) // Ensure valid memory access
        {
            temp = d_out[index] + d_out[index - stride];
        }
        __syncthreads();

        if (index >= stride) // Update output array
        {
            d_out[index] = temp;
        }
        __syncthreads();
    }
}