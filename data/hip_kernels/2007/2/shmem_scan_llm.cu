#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void shmem_scan(float* d_out, float* d_in) {
    extern __shared__ float sdata[];
    int idx = threadIdx.x;

    // Load input into shared memory
    sdata[idx] = d_in[idx];
    __syncthreads();

    // Optimize loop by using shared memory directly and avoiding redundant checks
    for (int interpre = 1; interpre < blockDim.x; interpre *= 2) {
        float temp = 0.0f;
        if (idx >= interpre) {
            temp = sdata[idx] + sdata[idx - interpre];
        }
        __syncthreads();
        if (idx >= interpre) {
            sdata[idx] = temp;
        }
        __syncthreads();
    }

    // Write result to global memory
    d_out[idx] = sdata[idx];
}