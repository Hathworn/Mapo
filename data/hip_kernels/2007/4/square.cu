#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square(float *d_in, float *d_out)
{
int index = threadIdx.x;
float f = d_in[index];
d_out[index] = f * f;
}