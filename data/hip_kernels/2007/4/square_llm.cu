#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Utilize shared memory for coalesced global memory access
__global__ void square(float *d_in, float *d_out)
{
    extern __shared__ float s_data[];

    int index = threadIdx.x;
    int global_index = blockIdx.x * blockDim.x + index;

    // Load data into shared memory
    s_data[index] = d_in[global_index];
    __syncthreads();

    // Perform computation
    float f = s_data[index];
    d_out[global_index] = f * f;
}