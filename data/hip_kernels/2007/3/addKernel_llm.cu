#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform addition if within bounds
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}