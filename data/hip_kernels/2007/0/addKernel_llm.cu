#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Use blockIdx.x and blockDim.x to process larger data with grid-stride loop
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    for (; i < size; i += blockDim.x * gridDim.x) {
        c[i] = a[i] + b[i];
    }
}