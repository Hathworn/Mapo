#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    // Use shared memory for pattern variable to reduce global memory writes
    __shared__ unsigned int pattern;
    pattern = 1;
    
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int* orig_ptr = ptr;

    unsigned long mask = 4;

    // Write pattern to the initial position
    *ptr = pattern;

    while(ptr < end_ptr){
        // Calculate new pointer position with mask and orig_ptr
        ptr = (unsigned int*) ( ((unsigned long)orig_ptr) | mask);
        
        if (ptr == orig_ptr){
            mask = mask << 1;
            continue;
        }
        if (ptr >= end_ptr) {
            break;
        }

        // Write pattern to the calculated position
        *ptr = pattern;

        // Double the pattern efficiently using bitwise operations
        pattern <<= 1;
        mask <<= 1;
    }
    return;
}