#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    // Check if the starting pointer for this block is within the valid range
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }
    
    // Use threadIdx.x to distribute work across threads within a block
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        ptr[i] = pattern;
    }

    return;
}