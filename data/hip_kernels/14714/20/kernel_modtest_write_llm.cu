#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    // Calculate the starting pointer for the current block
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check if the starting pointer goes beyond the end_ptr
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Cache block size in register
    unsigned int blockSizeInUInts = BLOCKSIZE / sizeof(unsigned int);
    
    // Using separate threads for the two tasks to enable parallelization
    unsigned int tid = threadIdx.x;
    unsigned int stride = blockDim.x;

    // Fill array with p1 at specified offsets in a strided loop
    for (unsigned int i = offset + tid; i < blockSizeInUInts; i += MOD_SZ * stride) {
        ptr[i] = p1;
    }

    // Fill array with p2 where the offsets don't match
    for (unsigned int i = tid; i < blockSizeInUInts; i += stride) {
        if (i % MOD_SZ != offset) {
            ptr[i] = p2;
        }
    }
}