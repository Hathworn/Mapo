#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    // Calculate the global thread index
    unsigned int idx = blockIdx.x * BLOCKSIZE + threadIdx.x * sizeof(unsigned long);
    unsigned long* ptr = (unsigned long*) (_ptr + idx);

    // Ensure ptr doesn't exceed end_ptr
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }

    // Write to memory using each thread to parallelize work
    ptr[0] = (unsigned long) &ptr[0];
}