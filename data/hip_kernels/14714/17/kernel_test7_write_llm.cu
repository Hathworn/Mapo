#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* start_ptr = (unsigned int*) _start_ptr;

    // Check if the starting pointer is within bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Utilize pointer arithmetic for better performance
    unsigned int* end_block_ptr = ptr + BLOCKSIZE / sizeof(unsigned int);

    // Use a while loop instead of for for slightly better performance
    while (ptr < end_block_ptr) {
        *ptr++ = *start_ptr++;
    }
}