#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel_move_inv_write function
__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    // Calculate block-based pointer
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Check bounds and exit early if needed
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Use loop unrolling to optimize the write operation
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i += 4) {
        ptr[i] = pattern;
        ptr[i + 1] = pattern;
        ptr[i + 2] = pattern;
        ptr[i + 3] = pattern;
    }

    return;
}