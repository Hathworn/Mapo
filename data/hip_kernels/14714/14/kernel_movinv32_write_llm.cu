#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset) {
    // Calculate the initial pointer and check boundary conditions
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Pre-calculate the block size in terms of unsigned int
    const unsigned int num_elements = BLOCKSIZE / sizeof(unsigned int);

    // Initialize variables for pattern manipulation
    unsigned int k = offset;
    unsigned int pat = pattern;

    // Unroll the loop to increase parallel execution efficiency
    #pragma unroll 4
    for (unsigned int i = 0; i < num_elements; i++) {
        ptr[i] = pat; // Write current pattern to memory
        k++; // Increment pattern position

        // Adjust pattern and reset `k` if necessary
        if (k >= 32) {
            k = 0; // Reset `k`
            pat = lb; // Set pattern to lower bound
        } else {
            pat = (pat << 1) | sval; // Shift pattern and apply sval
        }
    }
}