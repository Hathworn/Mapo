#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr) {
    unsigned int* orig_ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* ptr = orig_ptr;
    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);

    // Early exit if the block starts beyond end_ptr
    if (ptr >= (unsigned int*)end_ptr) return;

    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Initialize first value
    *ptr = pattern;

    // Loop through memory block and write patterns
    while (true) {
        ptr = (unsigned int*)(((unsigned long)orig_ptr) | mask);

        // Check for out-of-bounds or repeated address
        if (ptr >= block_end || ptr == orig_ptr) {
            mask <<= 1;
            if (ptr == orig_ptr) continue;
            else break;
        }

        *ptr = pattern;
        pattern <<= 1;
        mask <<= 1;
    }
}