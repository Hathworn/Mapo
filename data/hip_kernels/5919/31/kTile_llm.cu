#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    const uint numEls = tgtWidth * tgtHeight; // Precompute total elements to avoid computing every iteration

    for (uint i = idx; i < numEls; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight; // Use modulo to map target dimensions to source
        const uint srcX = x % srcWidth;
        tgt[i] = __ldg(&src[srcY * srcWidth + srcX]); // Use __ldg for read-only cache optimization
    }
}