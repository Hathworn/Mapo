#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    /* Optimize calculation of global thread index */
    hiprand_init(seed, tidx, 0, &state[tidx]);
}