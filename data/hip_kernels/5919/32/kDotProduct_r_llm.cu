#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numCols, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0; // Use register instead of shared memory for accumulation
    if (eidx < numCols) {
        for (; eidx < numElements; eidx += numCols) {
            sum += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = sum; // Store result in shared memory after loop

    __syncthreads();
    if (threadIdx.x < 128) { // Reduce synchronization points
        shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    }
    __syncthreads();
    if (threadIdx.x < 64) {
        shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    }
    __syncthreads();
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem; // Use volatile for warp-level reduction
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0]; // Write result to global memory
    }
}