#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stream(float *dA, float *dB, float *dC, float alpha, int N) {
    // Optimize by unrolling the loop to increase ILP (Instruction-Level Parallelism)
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = id; i < N; i += stride) {
        dA[i] = dB[i] + alpha * dC[i];
    }
}