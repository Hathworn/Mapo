#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bs(float *drand, float *dput, float *dcall, int n) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < n) {
        float c1 = 0.319381530f;
        float c2 = -0.356563782f;
        float c3 = 1.781477937f;
        float c4 = -1.821255978f;
        float c5 = 1.330274429f;

        float zero = 0.0f;
        float one = 1.0f;
        float two = 2.0f;
        float temp4 = 0.2316419f;

        float oneBySqrt2pi = 0.398942280f;

        float d1, d2;
        float phiD1, phiD2;
        float sigmaSqrtT;
        float KexpMinusRT;

        float inRand = drand[id];

        float S = S_LOWER_LIMIT * inRand + S_UPPER_LIMIT * (1.0f - inRand);
        float K = K_LOWER_LIMIT * inRand + K_UPPER_LIMIT * (1.0f - inRand);
        float T = T_LOWER_LIMIT * inRand + T_UPPER_LIMIT * (1.0f - inRand);
        float R = R_LOWER_LIMIT * inRand + R_UPPER_LIMIT * (1.0f - inRand);
        float sigmaVal = SIGMA_LOWER_LIMIT * inRand + SIGMA_UPPER_LIMIT * (1.0f - inRand);

        sigmaSqrtT = sigmaVal * sqrtf(T); // Use sqrtf for single precision

        d1 = (logf(S / K) + (R + sigmaVal * sigmaVal / two) * T) / sigmaSqrtT; // Use logf for single precision
        d2 = d1 - sigmaSqrtT;

        KexpMinusRT = K * expf(-R * T); // Use expf for single precision

        // Define a lambda to calculate the standard normal CDF
        auto phi = [&] (float X) -> float {
            float absX = fabsf(X); // Use fabsf for single precision
            float t = one / (one + temp4 * absX);
            float y = one - oneBySqrt2pi * expf(-X * X / two) * t * (c1 + t * (c2 + t * (c3 + t * (c4 + t * c5))));
            return (X < zero) ? (one - y) : y; // Ternary operator for conditional assignment
        };

        phiD1 = phi(d1);
        phiD2 = phi(d2);

        dcall[id] = S * phiD1 - KexpMinusRT * phiD2;

        phiD1 = phi(-d1); // Reuse the lambda for -d1
        phiD2 = phi(-d2); // Reuse the lambda for -d2

        dput[id] = KexpMinusRT * phiD2 - S * phiD1;
    }
}