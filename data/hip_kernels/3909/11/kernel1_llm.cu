#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel1(float *dW, float *dWcurr, int N) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure id is within bounds and copy data
    if (id < N) {
        dWcurr[id] = dW[id];
    }
    // Coalesce memory access by using stride loop
    id += blockDim.x * gridDim.x;
    while (id < N) {
        dWcurr[id] = dW[id];
        id += blockDim.x * gridDim.x;
    }
}