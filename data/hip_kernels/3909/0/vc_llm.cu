#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vc(float *dA, float *dB, int N) {
    // Calculate global thread index for current thread.
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Use if statement to check boundary and avoid branch divergence.
    if (id < N) {
        // Directly copy data from dB to dA for valid thread indices.
        dA[id] = dB[id];
    }
}