#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stream(float* dA, float* dB, float* dC, float alpha, int N) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < N) {
        // Use shared memory for faster access
        __shared__ float sharedB[BLOCK_SIZE];
        __shared__ float sharedC[BLOCK_SIZE];
        
        sharedB[threadIdx.x] = dB[id];
        sharedC[threadIdx.x] = dC[id];
        
        __syncthreads();  // Ensure all data is available

        dA[id] = sharedB[threadIdx.x] + alpha * sharedC[threadIdx.x];
    }
}
```
