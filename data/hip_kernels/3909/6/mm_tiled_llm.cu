#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mm_tiled(float *dA, float *dB, float *dC, int DIM, int N, int GPUN) {
    int it = blockIdx.y * 32;
    int jt = blockIdx.x * 32;
    int i = it + threadIdx.y;
    int j = jt + threadIdx.x;
    
    __shared__ float sA[32][32]; 
    __shared__ float sB[32][32];

    if (i * DIM + j < GPUN) {
        float sum = 0.0f;

        // Optimization: Reduce register usage by using variables for indexes
        for (int kt = 0; kt < DIM; kt += 32) {
            int rowA = it + threadIdx.y;
            int colA = kt + threadIdx.x;
            int rowB = kt + threadIdx.y;
            int colB = jt + threadIdx.x;

            // Load data into shared memory
            sA[threadIdx.y][threadIdx.x] = dA[rowA * DIM + colA];
            sB[threadIdx.y][threadIdx.x] = dB[rowB * DIM + colB];
            __syncthreads();

            // Compute using shared memory
            for (int k = 0; k < 32; k++) {
                sum += sA[threadIdx.y][k] * sB[k][threadIdx.x];
            }
            __syncthreads();
        }
        
        dC[i * DIM + j] = sum;
    }
}