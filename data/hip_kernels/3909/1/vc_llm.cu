#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vc(float *dA, float *dB, int N) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use __restrict__ to avoid pointer aliasing
    float *__restrict__ dA_loc = dA;
    float *__restrict__ dB_loc = dB;

    if (id < N) {
        // Minimize memory access by reading and writing directly
        dA_loc[id] = dB_loc[id];
    }
}