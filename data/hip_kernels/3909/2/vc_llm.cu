#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vc(float *dA, float *dB, int N) {
    // Calculate id only if there is work to be done
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= N) return;
    
    // Efficient memory access
    dA[id] = dB[id];
}