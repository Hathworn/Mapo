#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float phi(float X, float coeff1, float coeff2, float coeff3, float coeff4, float coeff5, float oneBySqrt2pi) {
    // Improved computation of phi using common expressions
    float temp4 = 0.2316419f;
    float absX = abs(X);
    float t = 1.0f / (1.0f + temp4 * absX);
    float expVar = exp(-X * X / 2.0f);
    float y = 1.0f - oneBySqrt2pi * expVar * t * (coeff1 + t * (coeff2 + t * (coeff3 + t * (coeff4 + t * coeff5))));
    return (X < 0.0f) ? (1.0f - y) : y;
}

__global__ void bs(float *drand, float *dput, float *dcall, int n) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) { 
        // Constants optimized using device function
        const float c1 = 0.319381530f;
        const float c2 = -0.356563782f;
        const float c3 = 1.781477937f;
        const float c4 = -1.821255978f;
        const float c5 = 1.330274429f;

        const float oneBySqrt2pi = 0.398942280f;
        const float zero = 0.0f, one = 1.0f, two = 2.0f;

        float d1, d2, phiD1, phiD2, sigmaSqrtT, KexpMinusRT;
        float inRand = drand[id];

        // Simplified randomization using one line for each
        float S = S_LOWER_LIMIT * inRand + S_UPPER_LIMIT * (1.0f - inRand);
        float K = K_LOWER_LIMIT * inRand + K_UPPER_LIMIT * (1.0f - inRand);
        float T = T_LOWER_LIMIT * inRand + T_UPPER_LIMIT * (1.0f - inRand);
        float R = R_LOWER_LIMIT * inRand + R_UPPER_LIMIT * (1.0f - inRand);
        float sigmaVal = SIGMA_LOWER_LIMIT * inRand + SIGMA_UPPER_LIMIT * (1.0f - inRand);

        sigmaSqrtT = sigmaVal * sqrt(T);
        d1 = (log(S / K) + (R + sigmaVal * sigmaVal / two) * T) / sigmaSqrtT;
        d2 = d1 - sigmaSqrtT;

        KexpMinusRT = K * exp(-R * T);

        // Utilizing device function to calculate phi
        phiD1 = phi(d1, c1, c2, c3, c4, c5, oneBySqrt2pi);
        phiD2 = phi(d2, c1, c2, c3, c4, c5, oneBySqrt2pi);

        dcall[id] = S * phiD1 - KexpMinusRT * phiD2;

        phiD1 = phi(-d1, c1, c2, c3, c4, c5, oneBySqrt2pi);
        phiD2 = phi(-d2, c1, c2, c3, c4, c5, oneBySqrt2pi);

        dput[id] = KexpMinusRT * phiD2 - S * phiD1;
    }
}