#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// ==== CONSTANTS ====

__constant__ unsigned int d_L;
__constant__ float d_dx;
__constant__ float d_dt;
__constant__ float d_mass;
__constant__ float d_g;
__constant__ unsigned int d_t;

__constant__ float d_scaling;
__constant__ int d_current_scene;

// ==== INITIALIZATION ====

__global__ void kernel_calculate_vertex_V(float3* vbo_ptr, float* d_ptr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < d_L && j < d_L) {
        // Optimize memory access by computing index once and reusing
        int index = d_L * i + j;
        vbo_ptr[index].y = d_scaling * fabsf(d_ptr[index]) - 0.005f * d_L;
    }
}