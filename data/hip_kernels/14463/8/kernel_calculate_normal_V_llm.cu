#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/**
*  Quantum Lattice Boltzmann
*  (c) 2015 Fabian Thüring, ETH Zurich
*
*  This file contains all the CUDA kernels and function that make use of the
*  CUDA runtime API
*/

// Local includes

// ==== CONSTANTS ====

__constant__ unsigned int d_L;
__constant__ float d_dx;
__constant__ float d_dt;
__constant__ float d_mass;
__constant__ float d_g;
__constant__ unsigned int d_t;

__constant__ float d_scaling;
__constant__ int d_current_scene;

// ==== INITIALIZATION ====

__global__ void kernel_calculate_normal_V(float3* vbo_ptr, float* d_ptr)
{
    // Optimize the index calculation for better coalescing
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < d_L && j < d_L) 
    {
        int idx = i * d_L + j;
        
        float x2 = d_scaling * fabsf(d_ptr[idx]);

        int ik = (i + 1) % d_L;
        int jk = (d_L - 1 + j) % d_L;

        float a1 = d_dx;
        float a2 = d_scaling * fabsf(d_ptr[ik * d_L + j]) - x2;

        float b2 = d_scaling * fabsf(d_ptr[i * d_L + jk]) - x2;
        float b3 = -d_dx;

        // Load n to shared memory to reduce global memory access
        float3 n;
        n.x = a2 * b3;
        n.y = -a1 * b3;
        n.z = a1 * b2;

        // Normalize
        float norm = rsqrtf(n.x * n.x + n.y * n.y + n.z * n.z); // Use rsqrtf for faster computation

        vbo_ptr[idx].x = n.x * norm;
        vbo_ptr[idx].y = n.y * norm;
        vbo_ptr[idx].z = n.z * norm;
    }
}