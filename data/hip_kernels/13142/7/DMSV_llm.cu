#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DMSV(float* M, float* V, float* R, int* addr, int N) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    if (tid < N) {
        // Load V into shared memory
        __shared__ float Vs[2 * LENGTH];
        if (tid < LENGTH) {
            for (int i = tid; i < 2 * N; i += BLOCK_SIZE) {
                Vs[i] = V[i];
            }
        }
        __syncthreads();

        // Calculate number of columns to process
        int numCols = (*addr - (intptr_t)&V[0]) / 4;
        float psum = 0.0;

        // Use shared memory for reduction
        for (int i = 0; i <= numCols; i++) {
            int vid = (int)Vs[i + N];
            psum += M[N * vid + bid * BLOCK_SIZE + tid] * Vs[i];
        }
        R[bid * BLOCK_SIZE + tid] = psum;
    }
}