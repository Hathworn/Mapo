#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n * c * batch_index;
    idx += m * nsample * batch_index;
    out += m * nsample * c * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Use shared memory for repeated access to reduce global memory access
    extern __shared__ float shared_points[];

    for (int j = index; j < m; j += stride) {
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            
            // Load points into shared memory
            for (int l = threadIdx.x; l < c; l += stride) {
                shared_points[threadIdx.x] = points[ii * c + l];
            }
            __syncthreads();

            // Copy from shared memory to output
            for (int l = 0; l < c; ++l) {
                out[j * nsample * c + k * c + l] = shared_points[l];
            }
        }
    }
}