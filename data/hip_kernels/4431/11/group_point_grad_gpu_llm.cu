#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    // Calculate the batch index, point set index and sample index using thread and block indices
    int i = blockIdx.x;
    int j = blockIdx.y;
    int k = threadIdx.x;

    if (i < b && j < m && k < nsample) {  // Ensure indices are within bounds
        int ii = idx[j * nsample + k];
        for (int l = 0; l < c; ++l) {
            atomicAdd(&grad_points[ii * c + l], grad_out[j * nsample * c + k * c + l]);
        }
    }
}