#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx, int *pts_cnt) {
    int batch_index = blockIdx.x;
    xyz1 += n * 3 * batch_index;
    xyz2 += m * 3 * batch_index;
    idx += m * nsample * batch_index;
    pts_cnt += m * batch_index;

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index >= m) return;  // Exit if thread index exceeds m

    float x2 = xyz2[index * 3 + 0];
    float y2 = xyz2[index * 3 + 1];
    float z2 = xyz2[index * 3 + 2];

    int cnt = 0;
    for (int k = 0; k < n; ++k) {
        if (cnt == nsample) break;
        
        float x1 = xyz1[k * 3 + 0];
        float y1 = xyz1[k * 3 + 1];
        float z1 = xyz1[k * 3 + 2];
        float d = sqrtf((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1));
        
        if (d < radius) {
            if (cnt == 0) {
                for (int l = 0; l < nsample; ++l)
                    idx[index * nsample + l] = k; // Initialize all indices to k
            }
            idx[index * nsample + cnt] = k;
            cnt += 1;
        }
    }
    pts_cnt[index] = cnt;
}