#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, float *dist, int *idx, float *val) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    idx += m * k * batch_index;
    val += m * k * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Each thread will process multiple elements in a stride loop
    for (int j = index; j < m; j += stride) {
        float *p_dist = dist + j * n;

        // selection sort for the first k elements
        for (int s = 0; s < k; ++s) {
            int min = s;

            // find the min using a more compact for loop
            for (int t = s + 1; t < n; ++t) {
                if (p_dist[t] < p_dist[min]) {
                    min = t;
                }
            }
            // update idx and val
            idx[j * k + s] = min;  // Simplified address calculation.
            val[j * k + s] = p_dist[min];

            // swap min-th and s-th element
            float tmp = p_dist[min];
            p_dist[min] = p_dist[s];
            p_dist[s] = tmp;
        }
    }
}