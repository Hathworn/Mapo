#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    // Thread indices for batch and points
    int batchIdx = blockIdx.x; // Assign each block to handle a batch
    int pointIdx = threadIdx.x; // Each thread handles a different point in the batch

    if (batchIdx < b && pointIdx < m) {
        const float *current_xyz1 = xyz1 + batchIdx * n * 3;
        const float *current_xyz2 = xyz2 + batchIdx * m * 3;
        int *current_idx = idx + batchIdx * m * nsample;
        
        float x2 = current_xyz2[pointIdx * 3 + 0];
        float y2 = current_xyz2[pointIdx * 3 + 1];
        float z2 = current_xyz2[pointIdx * 3 + 2];

        int cnt = 0;
        for (int k = 0; k < n; ++k) {
            if (cnt == nsample)
                break; // Break if nsample points are already selected

            float x1 = current_xyz1[k * 3 + 0];
            float y1 = current_xyz1[k * 3 + 1];
            float z1 = current_xyz1[k * 3 + 2];
            float d = max(sqrtf((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1)), 1e-20f);
            
            if (d < radius) {
                if (cnt == 0) {
                    for (int l = 0; l < nsample; ++l)
                        current_idx[pointIdx * nsample + l] = k; // Fill all with the first valid point
                }
                current_idx[pointIdx * nsample + cnt] = k;
                cnt += 1;
            }
        }
    }
}