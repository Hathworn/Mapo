#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    // Utilize blockIdx.x for parallel execution across batches
    int batch = blockIdx.x;
    int index = threadIdx.x;
    
    // Increment pointers based on batch number and thread index
    points += n * c * batch;
    idx += m * nsample * batch;
    out += m * nsample * c * batch;

    // Iterate over each sample and channel using parallel threads
    for (int j = index; j < m; j += blockDim.x) { // Parallelize outer loop
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            for (int l = 0; l < c; ++l) {
                // Copy data from points to out
                out[j * nsample * c + k * c + l] = points[ii * c + l];
            }
        }
    }
}