#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    // Calculate global thread indices
    int batch_index = blockIdx.x;
    int point_index = blockIdx.y * blockDim.y + threadIdx.y;
    int sample_index = threadIdx.x;

    // Ensure within bounds
    if (batch_index < b && point_index < m && sample_index < nsample) {
        int index = idx[point_index * nsample + sample_index];

        points += batch_index * n * c;
        out += batch_index * m * nsample * c + point_index * nsample * c + sample_index * c;

        // Copy the point data
        for (int l = 0; l < c; ++l) {
            out[l] = points[index * c + l];
        }
    }
}