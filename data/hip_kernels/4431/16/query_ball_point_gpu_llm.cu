#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    // Optimize thread and block index calculations
    int batch_index = blockIdx.x;
    int index = threadIdx.x;
    int stride = blockDim.x;

    // Offset pointers by batch index times size per batch
    xyz1 += n * 3 * batch_index;
    xyz2 += m * 3 * batch_index;
    idx += m * nsample * batch_index;

    // Parallelize over points in xyz2 using threads
    for (int j = index; j < m; j += stride) {
        float x2 = xyz2[j * 3 + 0];
        float y2 = xyz2[j * 3 + 1];
        float z2 = xyz2[j * 3 + 2];
        int cnt = 0;
        for (int k = 0; k < n; ++k) {
            if (cnt == nsample)
                break; // only pick the FIRST nsample points in the ball

            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            float d = max(sqrtf((x2-x1)*(x2-x1) + (y2-y1)*(y2-y1) + (z2-z1)*(z2-z1)), 1e-20f);

            if (d < radius) {
                if (cnt == 0) {
                    // set ALL indices to k, for valid (repeating) indices
                    for (int l = 0; l < nsample; ++l) {
                        idx[j * nsample + l] = k;
                    }
                }
                idx[j * nsample + cnt] = k;
                cnt++;
            }
        }
    }
}