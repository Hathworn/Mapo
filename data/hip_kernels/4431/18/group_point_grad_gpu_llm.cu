#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;

    // Loop over points handled by each thread using a stride approach
    for (int id = index; id < b * m; id += total_threads) {
        int batch = id / m;
        int j = id % m;

        const int *idx_ptr = idx + (batch * m + j) * nsample;
        const float *grad_out_ptr = grad_out + ((batch * m + j) * nsample * c);
        float *grad_points_ptr = grad_points + batch * n * c;

        for (int k = 0; k < nsample; ++k) {
            int ii = idx_ptr[k];
            for (int l = 0; l < c; ++l) {
                atomicAdd(&grad_points_ptr[ii * c + l], grad_out_ptr[k * c + l]); // Use atomicAdd for thread safety
            }
        }
    }
}