#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n*c*batch_index;
    idx += m*nsample*batch_index;
    out += m*nsample*c*batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Use shared memory to improve memory access speed
    extern __shared__ int shared_idx[];

    for (int j=index; j<m; j+=stride) {
        for (int k=0; k<nsample; ++k) {
            shared_idx[threadIdx.x * nsample + k] = idx[j*nsample + k];
        }
        __syncthreads(); // Synchronize threads to ensure all indices are loaded

        for (int k=0; k<nsample; ++k) {
            int ii = shared_idx[threadIdx.x * nsample + k];
            for (int l=0; l<c; ++l) {
                out[j*nsample*c+k*c+l] = points[ii*c+l];
            }
        }
        __syncthreads(); // Ensure all operations are done before moving to next for
    }
}