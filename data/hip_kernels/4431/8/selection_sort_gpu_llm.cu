#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    outi += m * n * batch_index;
    out += m * n * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Loop unrolling to improve memory access pattern
    for (int j = index; j < m; j += stride) {
        #pragma unroll 4
        for (int s = 0; s < n; ++s) {
            out[j * n + s] = dist[j * n + s];
            outi[j * n + s] = s;
        }
    }

    float *p_dist;
    for (int j = index; j < m; j += stride) {
        p_dist = out + j * n;
        // Optimized selection sort loop
        for (int s = 0; s < k; ++s) {
            int min = s;
            // Reduce the range of comparison
            for (int t = s + 1; t < n; ++t) {
                min = (p_dist[t] < p_dist[min]) ? t : min; 
            }
            if (min != s) {
                // Use a single swap operation
                float tmp = p_dist[s];
                p_dist[s] = p_dist[min];
                p_dist[min] = tmp;
                int tmpi = outi[j * n + s];
                outi[j * n + s] = outi[j * n + min];
                outi[j * n + min] = tmpi;
            }
        }
    }
}