#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    outi += m * n * batch_index;
    out += m * n * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Use shared memory for improved performance
    extern __shared__ float shared_dist[];

    // Copy from dist to shared memory
    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            shared_dist[j * n + s] = dist[j * n + s];
            outi[j * n + s] = s;
        }
    }
    __syncthreads();

    float *p_dist;
    for (int j = index; j < m; j += stride) {
        p_dist = shared_dist + j * n;
        // Selection sort for the first k elements
        for (int s = 0; s < k; ++s) {
            int min = s;
            // Find the min
            for (int t = s + 1; t < n; ++t) {
                if (p_dist[t] < p_dist[min]) {
                    min = t;
                }
            }
            // Swap min-th and i-th element
            if (min != s) {
                float tmp = p_dist[min];
                p_dist[min] = p_dist[s];
                p_dist[s] = tmp;
                int tmpi = outi[j * n + min];
                outi[j * n + min] = outi[j * n + s];
                outi[j * n + s] = tmpi;
            }
        }
    }

    // Copy sorted elements from shared memory back to global memory
    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            out[j * n + s] = shared_dist[j * n + s];
        }
    }
}
```
