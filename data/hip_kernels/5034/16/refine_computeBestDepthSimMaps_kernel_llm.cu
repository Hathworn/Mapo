#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float refineDepthSubPixel(const float3& depths, const float3& sims)
{
    float outDepth = -1.0f;

    float simM1 = sims.x;
    float simP1 = sims.z;
    float sim1 = sims.y;
    simM1 = (simM1 + 1.0f) / 2.0f;
    simP1 = (simP1 + 1.0f) / 2.0f;
    sim1 = (sim1 + 1.0f) / 2.0f;

    if ((simM1 > sim1) && (simP1 > sim1))
    {
        float dispStep = -((simP1 - simM1) / (2.0f * (simP1 + simM1 - 2.0f * sim1)));

        float floatDepthM1 = depths.x;
        float floatDepthP1 = depths.z;

        float b = (floatDepthP1 + floatDepthM1) / 2.0f;
        float a = b - floatDepthM1;

        outDepth = a * dispStep + b;
    }

    return outDepth;
}

__global__ void refine_computeBestDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float3* isims, int isims_p, float3* idpts, int idpts_p, int width, int height, float simThr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure within bounds for computation
    if ((x < width) && (y < height))
    {
        float3 depths = idpts[y * idpts_p + x];
        float3 sims = isims[y * isims_p + x];

        // Use ternary operators for clearer logic flow
        float outDepth = (sims.x < sims.y) ? depths.x : depths.y;
        float outSim = (sims.x < sims.y) ? sims.x : sims.y;

        // Compare with third value
        outDepth = (sims.z < outSim) ? depths.z : outDepth;
        outSim = (sims.z < outSim) ? sims.z : outSim;

        // Optimize depth refinement
        float refinedDepth = refineDepthSubPixel(depths, sims);
        if (refinedDepth > 0.0f)
        {
            outDepth = refinedDepth;
        }

        // Store results conditionally based on threshold
        osim[y * osim_p + x] = (outSim < simThr) ? outSim : 1.0f;
        odpt[y * odpt_p + x] = (outSim < simThr) ? outDepth : -1.0f;
    }
}