#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generateGaussian_kernel(float* og, float delta, int radius)
{
    // Use shared memory to optimize access
    extern __shared__ float shared_mem[];

    int x = threadIdx.x - radius;
    // Precompute constant part of the equation to be reused
    float coeff = 1.0f / (2 * delta * delta);
    int idx = threadIdx.x;

    // Store intermediate calculation in shared memory
    shared_mem[idx] = x * x * coeff;

    // Synchronize to ensure complete shared memory writes
    __syncthreads();

    // Perform the final math operation using shared memory
    og[idx] = __expf(-shared_mem[idx]);
}