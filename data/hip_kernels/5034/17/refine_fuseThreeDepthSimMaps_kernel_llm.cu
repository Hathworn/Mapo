#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void refine_fuseThreeDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float* isimLst, int isimLst_p, float* idptLst, int idptLst_p, float* isimAct, int isimAct_p, float* idptAct, int idptAct_p, int width, int height, float simThr)
{
    // Calculate the global thread indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure indices are within bounds
    if((x < width) && (y < height))
    {
        // Declare local arrays to store depths and similarities
        float dpts[3];
        float sims[3];

        // Load data from global memory to local arrays
        dpts[0] = odpt[y * odpt_p + x];
        dpts[1] = idptLst[y * idptLst_p + x];
        dpts[2] = idptAct[y * idptAct_p + x];

        sims[0] = osim[y * osim_p + x];
        sims[1] = isimLst[y * isimLst_p + x];
        sims[2] = isimAct[y * isimAct_p + x];

        // Initialize output depth and similarity
        float outDpt = dpts[0];
        float outSim = simThr;

        // Replace manual comparisons with a loop for optimization
        #pragma unroll
        for (int i = 1; i < 3; ++i) 
        {
            if (sims[i] < outSim) 
            {
                outDpt = dpts[i];
                outSim = sims[i];
            }
        }

        // Write results back to global memory
        osim[y * osim_p + x] = outSim;
        odpt[y * odpt_p + x] = outDpt;
    }
}