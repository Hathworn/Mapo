#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_dilateFPPlaneDepthMapXpYp_kernel(float* fpPlaneDepthMap, int fpPlaneDepthMap_p, float* maskMap, int maskMap_p, int width, int height, int xp, int yp, float fpPlaneDepth)
{
    // Compute thread's global x and y index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Precompute adjusted indices
    int adjusted_x = x + xp;
    int adjusted_y = y + yp;
    
    // Check if both the original and adjusted indices are within bounds
    if(adjusted_x >= 0 && adjusted_y >= 0 && adjusted_x < width && adjusted_y < height && x < width && y < height)
    {
        // Retrieve depth only once
        float depth = maskMap[y * maskMap_p + x];
        if(depth > 0.0f)
        {
            // Update the target depth map
            fpPlaneDepthMap[adjusted_y * fpPlaneDepthMap_p + adjusted_x] = fpPlaneDepth;
        }
    }
}