#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuComputeDistanceGlobal(float* A, int wA, float* B, int wB, int dim, float* AB) {

    // Declaration of the shared memory arrays As and Bs used to store the sub-matrix of A and B
    __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Block index
    int block_x = blockIdx.x;
    int block_y = blockIdx.y;

    // Other variables
    float ssd = 0;

    // Compute global indices
    int x_a = block_y * BLOCK_DIM + ty;
    int x_b = block_x * BLOCK_DIM + tx;

    int stride_a = BLOCK_DIM * wA;
    int stride_b = BLOCK_DIM * wB;

    for (int k = 0; k < dim; k += BLOCK_DIM) {

        // Load the matrices from device memory to shared memory
        shared_A[ty][tx] = (x_a < wA && k + tx < dim) ? A[x_a * dim + k + tx] : 0;
        shared_B[ty][tx] = (x_b < wB && k + ty < dim) ? B[x_b * dim + k + ty] : 0;

        // Synchronize within the block to ensure shared memory is ready
        __syncthreads();

        if (x_a < wA && x_b < wB) {
            for (int j = 0; j < BLOCK_DIM; ++j) {
                float diff = shared_A[ty][j] - shared_B[j][tx];
                ssd += diff * diff;
            }
        }

        // Synchronize to make sure that the preceding computation is done 
        __syncthreads();
    }

    // Write the result to global memory
    if (x_a < wA && x_b < wB) {
        AB[x_a * wB + x_b] = ssd;
    }
}