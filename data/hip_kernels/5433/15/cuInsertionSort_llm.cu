#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuInsertionSort(float *dist, long *ind, int width, int height, int k) {
    // Variables
    float *p_dist;
    long *p_ind;
    float curr_dist, max_dist;
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (xIndex < width) {        
        // Pointer shift, initialization
        p_dist = dist + xIndex;
        p_ind = ind + xIndex;
        max_dist = p_dist[0];
        p_ind[0] = 1;

        // Part 1 : sort first k elements
        for (int l = 1; l < k; l++) {
            int curr_row = l * width;
            curr_dist = p_dist[curr_row];
            
            if (curr_dist < max_dist) {
                int i = l - 1;
                while (i >= 0 && p_dist[i * width] > curr_dist) i--;  // Find the insertion point
                
                for (int j = l; j > i + 1; j--) {
                    p_dist[j * width] = p_dist[(j - 1) * width];
                    p_ind[j * width] = p_ind[(j - 1) * width];
                }
                
                p_dist[(i + 1) * width] = curr_dist;
                p_ind[(i + 1) * width] = l + 1;
            } else {
                p_ind[l * width] = l + 1;
            }
            
            max_dist = p_dist[curr_row];
        }

        // Part 2 : insert element in the first k sorted lines
        for (int l = k; l < height; l++) {
            curr_dist = p_dist[l * width];
            
            if (curr_dist < max_dist) {
                int i = k - 1;
                while (i >= 0 && p_dist[i * width] > curr_dist) i--;  // Find the insertion point
                
                for (int j = k - 1; j > i + 1; j--) {
                    p_dist[j * width] = p_dist[(j - 1) * width];
                    p_ind[j * width] = p_ind[(j - 1) * width];
                }
                
                p_dist[(i + 1) * width] = curr_dist;
                p_ind[(i + 1) * width] = l + 1;
                max_dist = p_dist[(k - 1) * width];
            }
        }
    }
}