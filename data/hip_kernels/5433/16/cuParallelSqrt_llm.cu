#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel for square root computation
__global__ void cuParallelSqrt(float *dist, int width, int k) {
    // Use a single dimension for linear indexing
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;  // Calculate the stride for all threads
    unsigned int totalElements = width * k;

    // Loop over all elements in the 1D flattened array with stride
    for (unsigned int i = index; i < totalElements; i += stride) {
        dist[i] = sqrt(dist[i]);
    }
}