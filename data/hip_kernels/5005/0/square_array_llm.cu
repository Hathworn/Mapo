#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if index is within range to prevent out-of-bounds access
    if (idx < N) {
        float value = a[idx]; // Load value from global memory
        a[idx] = value * value; // Store squared value to global memory
    }
}