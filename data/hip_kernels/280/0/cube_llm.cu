#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cube(float * d_out, float * d_in) {
    int index = blockDim.x * blockIdx.x + threadIdx.x; // Compute global index
    float f = d_in[index];
    d_out[index] = f * f * f;
}