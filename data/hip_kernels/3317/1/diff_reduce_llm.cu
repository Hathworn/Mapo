#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diff_reduce(double *dev_w, double *feat, double *pos, int feat_dim, int pos_dim, int par0, int par1, int n_patch)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i == j || i >= n_patch || j >= n_patch)
        return;

    double feat_dist = 0.0;
    double pos_dist = 0.0;
    int feat_offi = i * feat_dim;
    int feat_offj = j * feat_dim;
    int pos_offi = i * pos_dim;
    int pos_offj = j * pos_dim;

    // Unroll loop for better performance
    #pragma unroll
    for (int k = 0; k < feat_dim; k++) {
        double feat_i = feat[feat_offi + k];
        double feat_j = feat[feat_offj + k];
        feat_dist += (feat_i - feat_j) * (feat_i - feat_j);
    }

    // Unroll loop for better performance
    #pragma unroll
    for (int k = 0; k < pos_dim; k++) {
        double pos_i = pos[pos_offi + k];
        double pos_j = pos[pos_offj + k];
        pos_dist += (pos_i - pos_j) * (pos_i - pos_j);
    }

    // Utilize a single exp call using multiplication
    dev_w[i + j * n_patch] = exp(-(feat_dist / (feat_dim * par0 * par0) + pos_dist / (pos_dim * par1 * par1)));
}