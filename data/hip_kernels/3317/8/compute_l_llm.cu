#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_l(double *dev_w, int n_patch)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Pre-compute stride for efficiency
    int N = n_patch * n_patch;

    for (; tid < N; tid += stride) { // Use for-loop with pre-computed stride
        dev_w[tid] = ((tid % (n_patch + 1) == 0) ? 1.0 : 0.0) - dev_w[tid];
    }
}