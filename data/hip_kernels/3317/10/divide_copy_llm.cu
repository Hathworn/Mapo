#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void divide_copy(double *dest, const double *src, int length, const double divisor)
{
    // Calculate 1.0 / divisor once per block
    __shared__ double factor;
    if (threadIdx.x == 0) {
        factor = 1.0 / divisor;
    }
    __syncthreads(); // Ensure factor is loaded before any thread uses it

    // Calculate global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Loop through data array to handle larger arrays efficiently
    for (int index = tid; index < length; index += blockDim.x * gridDim.x) {
        dest[index] = src[index] * factor;
    }
}