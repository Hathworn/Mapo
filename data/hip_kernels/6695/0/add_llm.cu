#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 10

// Sum Arrays
__global__ void add(int *x, int *y, int *z) {
    int tID = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique thread ID
    if (tID < N) {
        z[tID] = x[tID] + y[tID]; // Perform vector addition
    }
}