#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize hiprandState initialization by utilizing threadIdx for parallel efficiency.
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate unique thread index
    hiprand_init(seed, idx, 0, &state[idx]); // Initialize state for each thread
}