#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NUMAR_NODURI 500
#define NUMAR_MUCHII 500
#define COST_MAXIM 1000000

typedef struct {
    int nod1;
    int nod2;
} Muchie;

typedef struct {
    int nodId;
    bool vizitat;
} Nod;

// Finds the cost of the road from start node to stop node
__device__ __host__ int CautareMuchie(Nod start, Nod stop, Muchie *muchii, int *costuri) {
    for (int i = 0; i < NUMAR_MUCHII; i++)
        if (muchii[i].nod1 == start.nodId && muchii[i].nod2 == stop.nodId)
            return costuri[i];

    return COST_MAXIM;
}

__global__ void Cauta_Nod(Nod *noduri, Muchie *muchii, int *costuri, int *costTemporal, int *costFinal) {
    int nod = threadIdx.x;
    
    // Load shared memory once
    __shared__ Nod localNoduri[NUMAR_NODURI];
    if (threadIdx.x < NUMAR_NODURI) {
        localNoduri[threadIdx.x] = noduri[threadIdx.x];
    }
    __syncthreads();
    
    if (!localNoduri[nod].vizitat) {
        localNoduri[nod].vizitat = true;
        for (int n = 0; n < NUMAR_NODURI; n++) {
            // Search for edge cost
            int cost = CautareMuchie(localNoduri[nod], localNoduri[n], muchii, costuri);

            // Use atomicMin to avoid race conditions
            if (cost < COST_MAXIM) {
                atomicMin(&costFinal[n], costTemporal[nod] + cost);
            }
        }
    }
}