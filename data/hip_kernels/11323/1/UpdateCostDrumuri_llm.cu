#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NUMAR_NODURI 500
#define NUMAR_MUCHII 500
#define COST_MAXIM 1000000

typedef struct
{
    int nod1;
    int nod2;
} Muchie;

typedef struct
{
    int nodId;
    bool vizitat;
} Nod;

// Optimize cost update path from start node to stop node
__global__ void UpdateCostDrumuri(Nod *noduri, int *costuriTemporale, int *costuriFinale)
{
    int nod = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (nod < NUMAR_NODURI) 
    {
        // Use local variable to minimize global memory access
        int costTemp = costuriTemporale[nod];
        
        if (costTemp > costuriFinale[nod])
        {
            costTemp = costuriFinale[nod];
            noduri[nod].vizitat = false;
        }
        
        costuriFinale[nod] = costTemp;
    }
}