#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_add(float* a, float* b, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for performance improvement
    __shared__ float sharedA[256];
    __shared__ float sharedB[256];
    
    if (i < n) {
        sharedA[threadIdx.x] = a[i];
        sharedB[threadIdx.x] = b[i];
        __syncthreads(); // Ensure all threads have written to shared memory

        // Perform addition from shared memory
        sharedA[threadIdx.x] += sharedB[threadIdx.x];
        
        // Write result back to global memory
        a[i] = sharedA[threadIdx.x];
    }
}