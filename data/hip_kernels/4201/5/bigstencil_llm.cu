#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bigstencil(int* in, int* out) {
    // Optimize: Use shared memory for faster data access.
    __shared__ int shared_in[1024]; // Adjust size based on blockDim.x.
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory.
    shared_in[threadIdx.x] = in[i];
    __syncthreads(); // Ensure all data is loaded before proceeding.
    
    // Perform computation.
    // Use shared memory data for faster read.
    out[i] = shared_in[threadIdx.x] + 2;
}