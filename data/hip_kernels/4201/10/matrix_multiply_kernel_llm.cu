#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    // Get the index for the current thread
    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < dim_mn) {
        double out = 0.0;
        
        // Use shared memory for input vector to optimize memory access
        extern __shared__ double s_vector_in[];
        
        if (threadIdx.x < dim_mn) {
            s_vector_in[threadIdx.x] = vector_in[threadIdx.x];
        }
        __syncthreads(); // Ensure all threads have loaded data before proceeding

        // Perform matrix-vector multiplication
        for (long j = 0; j < dim_mn; j++) {
            out += matrix[i * dim_mn + j] * s_vector_in[j];
        }

        // Store result in output vector
        vector_out[i] = out;
    }
}