#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(int* input, int* output) {
    // Load data into shared memory
    __shared__ int tmp[TPB];
    int tid = threadIdx.x;
    int offset = blockIdx.x * blockDim.x + tid;
    tmp[tid] = input[offset];
    __syncthreads();

    // Perform reduction with warp unrolling
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride)
            tmp[tid] += tmp[tid + stride];
        __syncthreads();
    }

    // Write the result for this block
    if (tid == 0) {
        output[blockIdx.x] = tmp[0];
    }
}