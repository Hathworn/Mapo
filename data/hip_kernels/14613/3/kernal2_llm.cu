#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernal2(int *A, int k, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Use loop unrolling to optimize memory access
    int stride = gridDim.x * blockDim.x;
    for (int idx = i; idx < numElements; idx += stride)
    {
        A[idx] = A[idx] * k;
    }
}