#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernal1(int *A, int *B, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Use strided access pattern to improve memory coalescing
    for (int idx = i; idx < numElements; idx += blockDim.x * gridDim.x)
    {
        B[idx] = A[idx] + B[idx];
    }
}