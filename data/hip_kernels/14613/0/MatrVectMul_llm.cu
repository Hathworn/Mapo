#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 15000

using namespace std;

// Optimized kernel function for matrix-vector multiplication
__global__ void MatrVectMul(int *d_c, int *d_a, int *d_b)
{
    // Shared memory for shared computation per block
    __shared__ int shared_b[N];
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (threadIdx.x < N) {
        shared_b[threadIdx.x] = d_b[threadIdx.x];
    }
    __syncthreads();

    if (i < N) {
        int sum = 0;

        // Loop unrolling technique for improved performance
        for (int k = 0; k < N; k += 4) {
            sum += d_a[i + k * N] * shared_b[k]
                 + d_a[i + (k + 1) * N] * shared_b[k + 1]
                 + d_a[i + (k + 2) * N] * shared_b[k + 2]
                 + d_a[i + (k + 3) * N] * shared_b[k + 3];
        }
        
        d_c[i] = sum;
    }
}