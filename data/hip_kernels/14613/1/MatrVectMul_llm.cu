#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrVectMul(int *d_c, int *d_a, int *d_b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        int sum = 0; // Use a local variable for accumulation to reduce memory access
        for (int k = 0; k < N; k++)
        {
            sum += d_a[i + k * N] * d_b[k];
        }
        d_c[i] = sum;
    }
}