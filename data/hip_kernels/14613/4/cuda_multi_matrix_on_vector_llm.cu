#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_multi_matrix_on_vector(int *matrix, int *vector, int *new_vector, int numElements) {
    __shared__ int cache[threadsPerBlock];
    const int idx = blockDim.x * blockIdx.x + threadIdx.x; // Global index
    const int tIdx = threadIdx.x; // Thread index
    const int k = (numElements + threadsPerBlock - 1) / threadsPerBlock; // Number of chunks based on threads per block

    // Optimize loop by minimizing data transfer and improving thread utilization
    for (int i = 0; i < k; i++) {
        if (tIdx + threadsPerBlock * i < numElements) {
            cache[tIdx] = vector[tIdx + threadsPerBlock * i]; // Load chunk of vector into shared memory
        }
        __syncthreads();

        int min = numElements - i * threadsPerBlock;
        min = min > threadsPerBlock ? threadsPerBlock : min; // Determine min to handle leftover elements

        if (idx < numElements) {
            int temp = 0; // Use temp variable to accumulate results
            for (int j = 0; j < min; j++) {
                temp += cache[j] * matrix[(i * threadsPerBlock + j) * numElements + idx];
            }
            atomicAdd(&new_vector[idx], temp); // Use atomicAdd to avoid race conditions
        }
        __syncthreads();
    }
}