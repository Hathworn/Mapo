#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecmabite(int *out, int *in, int threads, std::size_t size)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Use standard indexing
    if (idx < size / 2) // Ensure the index is within bounds
    {
        out[idx] = in[2 * idx]; // Simplify with computed index
    }
}