#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ifpairmabite(int *v, std::size_t size)
{
    // Calculate global thread ID
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within bounds
    if (tid < size) {
        // Check for even number and update value
        int val = v[tid];
        if ((val & 1) == 0) { // Use bitwise operation for even check
            v[tid] = val * 2;
        }
    }
}