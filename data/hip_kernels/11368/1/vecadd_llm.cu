#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecadd(int *v0, int *v1, std::size_t size)
{
    // calculate global thread index
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    // ensure thread index is within bounds
    if (tid < size) {
        v0[tid] += v1[tid];
    }
}