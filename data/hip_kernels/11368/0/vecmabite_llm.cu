#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecmabite(int *out, int *in, std::size_t size)
{
    // Compute global thread index to allow function to work for any launch configuration
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Prevent out-of-bounds memory access
    if (idx < size / 2)
    {
        out[idx] = in[2 * idx];
    }
}