#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    int32_t idx = blockIdx.x * N + threadIdx.x;
    int32_t stride = blockDim.x;
    
    // Loop unrolling for coalesced memory access optimization
    for (int32_t i = idx; i < blockIdx.x * N + N; i += stride) {
        output[i] = -col_sum[i - blockIdx.x * N];
    }
}