#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using block and thread identifiers to improve parallelism
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique id for each thread
    hiprand_init(seed, id, 0, &state[id]); // Initialize state based on unique id
}