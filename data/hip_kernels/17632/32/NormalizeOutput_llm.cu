#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Use loop unrolling for better performance
    for (; idx < num_elements; idx += stride) {
        int output_idx = idx * 3;
        to_normalize[output_idx] = batch_index;
        to_normalize[output_idx + 1] = class_index;
        to_normalize[output_idx + 2] = static_cast<int64_t>(original[idx]);
    }
}