#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread-safe atomic operation to divide accuracy
    atomicExch(accuracy, *accuracy / N);
}