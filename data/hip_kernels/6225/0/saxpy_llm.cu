#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX_VALUE 10

__global__ void saxpy(float *X, float *Y, float *Z, int A, int N)
{
    // Calculate global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Use strided loop to allow one kernel launch for entire array
    for (; i < N; i += gridDim.x * blockDim.x) {
        Z[i] = A * X[i] + Y[i];
    }
}