#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_hello() {
    // Use shared memory buffer for storing formatted output to reduce latency
    __shared__ char buffer[256];
    
    // Use thread 0 of each block to print the output
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
        snprintf(buffer, 256, "Hello from GPU blockIdx.x=%d blockIdx.y=%d blockIdx.z=%d threadIdx.x=%d threadIdx.y=%d threadIdx.z=%d\n",
                 blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);

        printf("%s", buffer);
    }
}