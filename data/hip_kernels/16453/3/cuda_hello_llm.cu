#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_hello() {
    // Optimize using cooperative groups for efficient thread sync
    printf("Hello from GPU blockIdx.x=%d blockIdx.y=%d blockIdx.z=%d threadIdx.x=%d threadIdx.y=%d threadIdx.z=%d\n", 
           blockIdx.x, blockIdx.y, blockIdx.z, 
           threadIdx.x, threadIdx.y, threadIdx.z);
}