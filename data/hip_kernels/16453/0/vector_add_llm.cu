#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 10000000
#define MAX_ERR 1e-6

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;  // Calculate global thread index
    int stride = blockDim.x * gridDim.x;                // Calculate total stride across all blocks

    for(int i = index; i < n; i += stride){
        out[i] = a[i] + b[i];
    }
}