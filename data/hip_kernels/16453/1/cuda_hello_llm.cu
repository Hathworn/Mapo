#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use __restrict__ to hint compiler for no aliasing
__global__ void cuda_hello() {
    // Compute global thread index for efficiency
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello from GPU globalIdx=%d blockIdx.x=%d blockIdx.y=%d blockIdx.z=%d threadIdx.x=%d threadIdx.y=%d threadIdx.z=%d\n", globalIdx, blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
}