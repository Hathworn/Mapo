#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Use loop to handle multiple elements per thread, improving resource utilization
    for (; i < numElements; i += blockDim.x * gridDim.x)
    {
        C[i] = A[i] + B[i];
    }
}