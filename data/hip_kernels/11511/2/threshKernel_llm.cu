#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel with shared memory
__global__ void threshKernel(unsigned char * image, unsigned char* moddedimage, int size, int threshold)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Launch more threads than necessary to check bounds
    if (i >= size) return;

    // Use conditional operator for a concise threshold operation
    moddedimage[i] = (image[i] > threshold) ? 255 : 0;
}