#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Get global thread index for larger arrays
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure we do not exceed array bounds
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}
```
