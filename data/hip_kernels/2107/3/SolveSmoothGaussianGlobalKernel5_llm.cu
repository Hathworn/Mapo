#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SolveSmoothGaussianGlobalKernel5(float* u, float* v, float* bku, float* bkv, int width, int height, int stride, float *outputu, float *outputv, float *outputbku, float* outputbkv)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix >= width || iy >= height) return;

    const int pos = ix + iy * stride;
    float sumu = 0.0f;
    float sumv = 0.0f;

    __shared__ float w[25];
    if (threadIdx.x < 5 && threadIdx.y < 5) {
        const float local_w[25] = { 0.0037, 0.0147, 0.0256, 0.0147, 0.0037,
                                    0.0147, 0.0586, 0.0952, 0.0586, 0.0147,
                                    0.0256, 0.0952, 0.1502, 0.0952, 0.0256,
                                    0.0147, 0.0586, 0.0952, 0.0586, 0.0147,
                                    0.0037, 0.0147, 0.0256, 0.0147, 0.0037 };
        w[threadIdx.y * 5 + threadIdx.x] = local_w[threadIdx.y * 5 + threadIdx.x];
    }
    __syncthreads(); // Ensure all threads have loaded the weights

    for (int j = 0; j < 5; j++) {
        int row = iy + j - 2; // Precompute row index
        if (row >= 0 && row < height) { // Check the row boundary once
            for (int i = 0; i < 5; i++) {
                int col = ix + i - 2; // Precompute column index
                if (col >= 0 && col < width) { // Check boundary conditions
                    int idx = col + stride * row;
                    sumu += w[j * 5 + i] * u[idx];
                    sumv += w[j * 5 + i] * v[idx];
                }
            }
        }
    }

    outputu[pos] = sumu;
    outputv[pos] = sumv;
    outputbku[pos] = bku[pos] + u[pos] - sumu;
    outputbkv[pos] = bkv[pos] + v[pos] - sumv;
}