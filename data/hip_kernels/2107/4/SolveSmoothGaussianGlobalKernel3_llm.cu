#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SolveSmoothGaussianGlobalKernel3(float* u, float* v, float* bku, float* bkv, int width, int height, int stride, float* outputu, float* outputv, float* outputbku, float* outputbkv)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;
    const int pos = ix + iy * stride;

    if (ix >= width || iy >= height) return;
    
    // Preload weights into shared memory for faster access
    __shared__ float w[9];
    if (threadIdx.x < 9) {
        w[threadIdx.x] = (threadIdx.x == 1 || threadIdx.x == 3 || threadIdx.x == 5 || threadIdx.x == 7) ? 0.1667f : (threadIdx.x == 4) ? 0.3333f : 0.0f;
    }
    __syncthreads();

    float sumu = 0.0f;
    float sumv = 0.0f;

    // Unrolling loops for better performance
#pragma unroll
    for (int j = -1; j <= 1; j++) {
#pragma unroll
        for (int i = -1; i <= 1; i++) {
            int col = ix + i;
            int row = iy + j;
            if (col >= 0 && col < width && row >= 0 && row < height) {
                int index = (j + 1) * 3 + (i + 1);
                sumu += w[index] * u[col + stride * row];
                sumv += w[index] * v[col + stride * row];
            }
        }
    }
    outputu[pos] = sumu;
    outputv[pos] = sumv;
    outputbku[pos] = bku[pos] + u[pos] - sumu;
    outputbkv[pos] = bkv[pos] + v[pos] - sumv;
}