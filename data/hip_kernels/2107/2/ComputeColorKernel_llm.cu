#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeColorKernel(float *u, float *v, int width, int height, int stride, float3 *uvRGB, float flowscale) {
    int r = blockIdx.y * blockDim.y + threadIdx.y;  // current row
    int c = blockIdx.x * blockDim.x + threadIdx.x;  // current column

    if (r < height && c < width) {
        int pos = c + stride * r;
        float du = u[pos] / flowscale;
        float dv = v[pos] / flowscale;

        int ncols = 55;
        float rad = sqrtf(du * du + dv * dv);
        float a = atan2f(-dv, -du) / 3.14159f;
        float fk = (a + 1) * 0.5f * ((float)ncols - 1);
        int k0 = __float2int_rd(fk);  // fast floor alternative
        int k1 = k0 + 1;
        if (k1 >= ncols) {
            k1 = 0;
        }
        float f = fk - (float)k0;

        __shared__ float colorwheelR[55], colorwheelG[55], colorwheelB[55];
        if (threadIdx.x == 0 && threadIdx.y == 0) {
            float tempR[55] = {255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
                               255, 213, 170, 128, 85, 43, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                               0, 0, 0, 0, 0, 0, 0, 19, 39, 58, 78, 98, 117, 137, 156,
                               176, 196, 215, 235, 255, 255, 255, 255, 255, 255};
            float tempG[55] = {0, 17, 34, 51, 68, 85, 102, 119, 136, 153, 170, 187, 204, 221, 238,
                               255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 232, 209, 186, 163,
                               140, 116, 93, 70, 47, 24, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                               0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
            float tempB[55] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
                               0, 0, 0, 0, 0, 0, 0, 63, 127, 191, 255, 255, 255, 255, 255,
                               255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255, 255,
                               255, 255, 255, 255, 255, 213, 170, 128, 85, 43};
            for (int i = 0; i < 55; i++) {
                colorwheelR[i] = tempR[i];
                colorwheelG[i] = tempG[i];
                colorwheelB[i] = tempB[i];
            }
        }
        __syncthreads();

        float colR = (1 - f) * (colorwheelR[k0] / 255.0f) + f * (colorwheelR[k1] / 255.0f);
        float colG = (1 - f) * (colorwheelG[k0] / 255.0f) + f * (colorwheelG[k1] / 255.0f);
        float colB = (1 - f) * (colorwheelB[k0] / 255.0f) + f * (colorwheelB[k1] / 255.0f);

        if (rad <= 1) {
            colR = 1 - rad * (1 - colR);
            colG = 1 - rad * (1 - colG);
            colB = 1 - rad * (1 - colB);
        } else {
            colR *= 0.75f;
            colG *= 0.75f;
            colB *= 0.75f;
        }

        uvRGB[pos] = make_float3(colB, colG, colR);
    }
}