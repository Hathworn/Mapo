#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SolveSmoothMedianGlobalKernel3(float* u, float* v, float* bku, float* bkv, int width, int height, int stride, float *outputu, float *outputv, float *outputbku, float* outputbkv)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix >= width || iy >= height) return;  // Exit early if out of bounds

    const int pos = ix + iy * stride;
    float mu[9], mv[9];

    // Load 3x3 neighborhood
    #pragma unroll  // Unroll loops for performance
    for (int j = 0; j < 3; j++) {
        for (int i = 0; i < 3; i++) {
            int col = ix + i - 1;
            int row = iy + j - 1;
            int index = j * 3 + i;

            if (col < 0) col = 0;  // Clamp to border
            if (col >= width) col = width - 1;
            if (row < 0) row = 0;
            if (row >= height) row = height - 1;

            mu[index] = u[col + stride * row];
            mv[index] = v[col + stride * row];
        }
    }

    // Sort using selection sort - Optimized
    #pragma unroll  // Unroll loops for performance
    for (int j = 0; j < 9; j++) {
        for (int i = j + 1; i < 9; i++) {
            if (mu[j] > mu[i]) {
                float tmp = mu[j];
                mu[j] = mu[i];
                mu[i] = tmp;
            }
            if (mv[j] > mv[i]) {
                float tmp = mv[j];
                mv[j] = mv[i];
                mv[i] = tmp;
            }
        }
    }

    // Compute output values
    outputu[pos] = mu[4];
    outputv[pos] = mv[4];
    outputbku[pos] = bku[pos] + u[pos] - mu[4];
    outputbkv[pos] = bkv[pos] + v[pos] - mv[4];
}