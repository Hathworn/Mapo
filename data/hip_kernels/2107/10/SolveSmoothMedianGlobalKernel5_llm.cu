#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SolveSmoothMedianGlobalKernel5(float* u, float* v, float* bku, float* bkv, int width, int height, int stride, float *outputu, float *outputv, float *outputbku, float* outputbkv)
{
    const int ix = threadIdx.x + blockIdx.x * blockDim.x;
    const int iy = threadIdx.y + blockIdx.y * blockDim.y;

    const int pos = ix + iy * stride;

    if (ix >= width || iy >= height) return;

    float mu[25] = { 0.0f };
    float mv[25] = { 0.0f };

    #pragma unroll
    for (int j = 0; j < 5; j++) {
        #pragma unroll
        for (int i = 0; i < 5; i++) {
            int col = ix + i - 2;
            int row = iy + j - 2;
            int clamped_col = min(max(col, 0), width - 1);
            int clamped_row = min(max(row, 0), height - 1);

            mu[j * 5 + i] = u[clamped_col + stride * clamped_row];
            mv[j * 5 + i] = v[clamped_col + stride * clamped_row];
        }
    }

    // Optimize sorting using insertion sort for small array
    float tmpu, tmpv;
    #pragma unroll
    for (int j = 1; j < 25; j++) {
        tmpu = mu[j];
        tmpv = mv[j];
        int i = j - 1;
        while (i >= 0 && mu[i] > tmpu) {
            mu[i + 1] = mu[i];
            i--;
        }
        mu[i + 1] = tmpu;

        i = j - 1;
        while (i >= 0 && mv[i] > tmpv) {
            mv[i + 1] = mv[i];
            i--;
        }
        mv[i + 1] = tmpv;
    }

    outputu[pos] = mu[12];
    outputv[pos] = mv[12];
    outputbku[pos] = bku[pos] + u[pos] - mu[12];
    outputbkv[pos] = bkv[pos] + v[pos] - mv[12];
}