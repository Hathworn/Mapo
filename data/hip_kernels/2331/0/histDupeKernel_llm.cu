#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histDupeKernel(const float* data1, const float* data2, const float* confidence1, const float* confidence2, int* ids1, int* ids2, int* results_id1, int* results_id2, float* results_similarity, int* result_count, const int N1, const int N2, const int max_results) {

    const unsigned int thread = threadIdx.x;
    const unsigned int block = blockIdx.x;
    const unsigned int block_size = blockDim.x;
    const unsigned int block_start = block_size * block;
    const unsigned int index = block_start + thread;

    float conf = (index < N1) ? confidence1[index] : 0.0f; // Guard against out-of-bounds access
    int id = (index < N1) ? ids1[index] : -1;

    __shared__ float hists[128 * 64];
    if (index < N1) { // Guard against filling shared memory beyond bounds
        for (unsigned int i = 0; i < 64; i++) {
            hists[i * 128 + thread] = data1[(block_start + i) * 128 + thread];
            hists[i * 128 + thread + 64] = data1[(block_start + i) * 128 + 64 + thread];
        }
    }

    __shared__ float other[128];

    for (unsigned int i = 0; i < N2 && *result_count < max_results; i++) {

        if (thread < 128) { // Let only threads within bounds contribute
            other[thread] = data2[i * 128 + thread];
            other[thread + 64] = data2[i * 128 + thread + 64];
        }

        __syncthreads();

        if (index < N1) { // Ensure thread is within bounds before computation
            float d = 0;
            for (unsigned int k = 0; k < 128; k++) {
                d += fabsf(hists[thread * 128 + k] - other[k]);
            }
            d = 1 - (d / 8);

            int other_id = ids2[i];

            if (other_id != id && d > fmaxf(conf, confidence2[i])) {
                int result_index = atomicAdd(result_count, 1);
                if (result_index < max_results) {
                    results_similarity[result_index] = d;
                    results_id1[result_index] = id;
                    results_id2[result_index] = other_id;
                }
            }
        }

        __syncthreads();
    }
}