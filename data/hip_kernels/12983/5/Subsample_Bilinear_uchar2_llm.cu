#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subsample_Bilinear_uchar2(hipTextureObject_t uchar2_tex, uchar2 *dst, int dst_width, int dst_height, int dst_pitch2, int src_width, int src_height) 
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width) 
    {
        float hscale = __fdividef(src_width, dst_width); // Use fast division
        float vscale = __fdividef(src_height, dst_height); // Use fast division
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;
        
        // Precompute shared weights for efficiency
        float wh = __saturatef(0.5f * (hscale - 1.0f));
        float wv = __saturatef(0.5f * (vscale - 1.0f));
        
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);
        
        // Load all textures once for better memory coalescing
        uchar2 c0 = tex2D<uchar2>(uchar2_tex, xi - dx, yi - dy);
        uchar2 c1 = tex2D<uchar2>(uchar2_tex, xi + dx, yi - dy);
        uchar2 c2 = tex2D<uchar2>(uchar2_tex, xi - dx, yi + dy);
        uchar2 c3 = tex2D<uchar2>(uchar2_tex, xi + dx, yi + dy);

        int2 uv;
        uv.x = ((int)c0.x + (int)c1.x + (int)c2.x + (int)c3.x + 2) >> 2;
        uv.y = ((int)c0.y + (int)c1.y + (int)c2.y + (int)c3.y + 2) >> 2;
        
        dst[yo * dst_pitch2 + xo] = make_uchar2((unsigned char)uv.x, (unsigned char)uv.y);
    }
}