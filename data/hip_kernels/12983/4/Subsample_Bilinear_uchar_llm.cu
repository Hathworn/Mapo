#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subsample_Bilinear_uchar(hipTextureObject_t uchar_tex, unsigned char *dst, int dst_width, int dst_height, int dst_pitch, int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensuring calculations are done only within bounds
    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / dst_width;
        float vscale = (float)src_height / dst_height;

        // Calculate source image coordinates
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;

        // Pre-calculate scaled halfweights for bilinear filtering
        float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
        float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);

        // Sample texture at calculated positions
        int y0 = tex2D<unsigned char>(uchar_tex, xi - dx, yi - dy);
        int y1 = tex2D<unsigned char>(uchar_tex, xi + dx, yi - dy);
        int y2 = tex2D<unsigned char>(uchar_tex, xi - dx, yi + dy);
        int y3 = tex2D<unsigned char>(uchar_tex, xi + dx, yi + dy);

        // Store result with optimized average calculation
        dst[yo * dst_pitch + xo] = (unsigned char)((y0 + y1 + y2 + y3 + 2) >> 2);
    }
}