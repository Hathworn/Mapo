#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Thumbnail_uchar(hipTextureObject_t uchar_tex, int *histogram, int src_width, int src_height)
{
    // Calculate the global index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check boundary conditions
    if (y < src_height && x < src_width)
    {
        // Optimize memory access by using a register for pixel
        unsigned char pixel = tex2D<unsigned char>(uchar_tex, x, y);
        
        // Use atomic operations safely
        atomicAdd(&histogram[pixel], 1);
    }
}