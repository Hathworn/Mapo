#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Thumbnail_ushort(hipTextureObject_t ushort_tex, int *histogram, int src_width, int src_height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < src_height && x < src_width)
    {
        // Improve texture fetching for 2D cache locality
        unsigned short pixel = (tex2D<unsigned short>(ushort_tex, x + 0.5f, y + 0.5f) + 128) >> 8;
        
        // Use shared memory for histogram to reduce global atomic operations
        __shared__ int shared_histogram[256];
        
        if (threadIdx.x < 256) 
            shared_histogram[threadIdx.x] = 0;
        
        __syncthreads();
        
        atomicAdd(&shared_histogram[pixel], 1);
        
        __syncthreads();
        
        // Accumulate results back to global memory
        if (threadIdx.x < 256)
            atomicAdd(&histogram[threadIdx.x], shared_histogram[threadIdx.x]);
    }
}