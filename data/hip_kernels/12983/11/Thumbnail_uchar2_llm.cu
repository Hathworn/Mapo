#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Thumbnail_uchar2(hipTextureObject_t uchar2_tex, int *histogram, int src_width, int src_height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (y < src_height && x < src_width)
    {
        // Load pixel data
        uchar2 pixel = tex2D<uchar2>(uchar2_tex, x, y);
        // Utilize shared memory for histogram update
        __shared__ int local_histogram[512];
        
        int thread_id = threadIdx.y * blockDim.x + threadIdx.x;
        if (thread_id < 512)
        {
            local_histogram[thread_id] = 0;
        }
        __syncthreads();
        
        atomicAdd(&local_histogram[pixel.x], 1);
        atomicAdd(&local_histogram[256 + pixel.y], 1);
        __syncthreads();

        if (thread_id < 512)
        {
            atomicAdd(&histogram[thread_id], local_histogram[thread_id]);
        }
    }
}