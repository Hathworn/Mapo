#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Overlay_Cuda(int x_position, int y_position, unsigned char* main, int main_linesize, unsigned char* overlay, int overlay_linesize, int overlay_w, int overlay_h, unsigned char* overlay_alpha, int alpha_linesize, int alpha_adj_x, int alpha_adj_y) 
{
    // Calculate the global x and y coordinates of the thread
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Early return if thread is outside the overlay region
    if (x < x_position || x >= x_position + overlay_w || y < y_position || y >= y_position + overlay_h) {
        return;
    }

    // Calculate overlay coordinates
    int overlay_x = x - x_position;
    int overlay_y = y - y_position;

    // Calculate alpha value if alpha_linesize is non-zero
    float alpha = 1.0f;
    if (alpha_linesize) {
        alpha = overlay_alpha[alpha_adj_x * overlay_x + alpha_adj_y * overlay_y * alpha_linesize] / 255.0f;
    }

    // Compute the new pixel value for the main image
    int main_idx = x + y * main_linesize;
    main[main_idx] = alpha * overlay[overlay_x + overlay_y * overlay_linesize] + (1.0f - alpha) * main[main_idx];
}