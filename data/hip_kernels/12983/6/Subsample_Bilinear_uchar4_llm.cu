#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subsample_Bilinear_uchar4(hipTextureObject_t uchar4_tex, uchar4 *dst, int dst_width, int dst_height, int dst_pitch, int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        // Calculate scales and fractional positions
        float hscale = (float)src_width / dst_width;
        float vscale = (float)src_height / dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;
        
        // Simplify weight calculations using inline expressions
        float dx = fminf(fmaxf(0.5f * (hscale - 1.0f), 0.0f), 1.0f) / 0.5f;
        float dy = fminf(fmaxf(0.5f * (vscale - 1.0f), 0.0f), 1.0f) / 0.5f;
        
        // Fetch texture data with reduced dependency on intermediate variables
        uchar4 c0 = tex2D<uchar4>(uchar4_tex, xi - dx, yi - dy);
        uchar4 c1 = tex2D<uchar4>(uchar4_tex, xi + dx, yi - dy);
        uchar4 c2 = tex2D<uchar4>(uchar4_tex, xi - dx, yi + dy);
        uchar4 c3 = tex2D<uchar4>(uchar4_tex, xi + dx, yi + dy);

        // Sum and average the four texels
        int4 res;
        res.x = ((int)c0.x + (int)c1.x + (int)c2.x + (int)c3.x + 2) >> 2;
        res.y = ((int)c0.y + (int)c1.y + (int)c2.y + (int)c3.y + 2) >> 2;
        res.z = ((int)c0.z + (int)c1.z + (int)c2.z + (int)c3.z + 2) >> 2;
        res.w = ((int)c0.w + (int)c1.w + (int)c2.w + (int)c3.w + 2) >> 2;

        // Store the result back to the destination array
        dst[yo * dst_pitch + xo] = make_uchar4(
            (unsigned char)res.x, (unsigned char)res.y, (unsigned char)res.z, (unsigned char)res.w);
    }
}