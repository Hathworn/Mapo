#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "Subsample_Bilinear_uchar4.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    hipTextureObject_t uchar4_tex = 1;
uchar4 *dst = NULL;
hipMalloc(&dst, XSIZE*YSIZE);
int dst_width = XSIZE;
int dst_height = YSIZE;
int dst_pitch = 2;
int src_width = XSIZE;
int src_height = YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
Subsample_Bilinear_uchar4<<<gridBlock, threadBlock>>>(uchar4_tex,dst,dst_width,dst_height,dst_pitch,src_width,src_height);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
Subsample_Bilinear_uchar4<<<gridBlock, threadBlock>>>(uchar4_tex,dst,dst_width,dst_height,dst_pitch,src_width,src_height);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
Subsample_Bilinear_uchar4<<<gridBlock, threadBlock>>>(uchar4_tex,dst,dst_width,dst_height,dst_pitch,src_width,src_height);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}