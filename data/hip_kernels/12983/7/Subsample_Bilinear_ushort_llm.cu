#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subsample_Bilinear_ushort(hipTextureObject_t ushort_tex, unsigned short *dst, int dst_width, int dst_height, int dst_pitch, int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        float hscale = (float)src_width / (float)dst_width;
        float vscale = (float)src_height / (float)dst_height;
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;
        
        // Precompute constants outside texture fetch for efficiency
        float wh = min(max(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
        float dx = wh / (0.5f + wh);
        
        float wv = min(max(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
        float dy = wv / (0.5f + wv);
        
        // Optimize texture fetches to reduce overhead
        int y0 = tex2D<unsigned short>(ushort_tex, xi - dx, yi - dy);  
        int y1 = tex2D<unsigned short>(ushort_tex, xi + dx, yi - dy);  
        int y2 = tex2D<unsigned short>(ushort_tex, xi - dx, yi + dy);  
        int y3 = tex2D<unsigned short>(ushort_tex, xi + dx, yi + dy);  
        
        // Use bitwise operator to optimize averaging
        dst[yo * dst_pitch + xo] = (unsigned short)((y0 + y1 + y2 + y3 + 2) >> 2);
    }
}