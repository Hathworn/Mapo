#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Thumbnail_ushort2(hipTextureObject_t ushort2_tex, int *histogram, int src_width, int src_height)
{
    // Calculate thread's absolute position in 2D space
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Use shared memory for histogram accumulation to reduce global memory access
    __shared__ int shared_histogram[512];
    
    // Initialize shared histogram in parallel
    if (threadIdx.x < 512)
        shared_histogram[threadIdx.x] = 0;
    __syncthreads();

    // Ensure pixel location is valid and within image bounds
    if (y < src_height && x < src_width)
    {
        // Texture fetch
        ushort2 pixel = tex2D<ushort2>(ushort2_tex, x, y);
        
        // Atomic update to shared histogram
        atomicAdd(&shared_histogram[(pixel.x + 128) >> 8], 1);
        atomicAdd(&shared_histogram[256 + ((pixel.y + 128) >> 8)], 1);
    }
    __syncthreads();

    // Transfer shared histogram data to global memory
    if (threadIdx.x < 512)
        atomicAdd(&histogram[threadIdx.x], shared_histogram[threadIdx.x]);
}