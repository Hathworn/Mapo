#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subsample_Bilinear_ushort2(hipTextureObject_t ushort2_tex, ushort2 *dst, int dst_width, int dst_height, int dst_pitch2, int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        // Precompute scale factors outside the loop for efficiency
        float hscale = __fdividef(src_width, dst_width);
        float vscale = __fdividef(src_height, dst_height);
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;
        
        // Precompute weights using fminf and fmaxf for better performance
        float wh = fminf(fmaxf(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
        float wv = fminf(fmaxf(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
        
        float dx = __fdividef(wh, (0.5f + wh));
        float dy = __fdividef(wv, (0.5f + wv));
        
        // Read texture data with fewer texture fetches
        ushort2 c0 = tex2D<ushort2>(ushort2_tex, xi - dx, yi - dy);
        ushort2 c1 = tex2D<ushort2>(ushort2_tex, xi + dx, yi - dy);
        ushort2 c2 = tex2D<ushort2>(ushort2_tex, xi - dx, yi + dy);
        ushort2 c3 = tex2D<ushort2>(ushort2_tex, xi + dx, yi + dy);
        
        int2 uv;
        // Use bitwise operations for efficient averaging
        uv.x = ((int)c0.x + (int)c1.x + (int)c2.x + (int)c3.x + 2) >> 2;
        uv.y = ((int)c0.y + (int)c1.y + (int)c2.y + (int)c3.y + 2) >> 2;
        
        dst[yo * dst_pitch2 + xo] = make_ushort2((unsigned short)uv.x, (unsigned short)uv.y);
    }
}