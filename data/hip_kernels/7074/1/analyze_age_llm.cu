#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
cudaStructTest
testing/optimizing how to access/manipulate/return
structures in cuda.
*/

#define N 30
#define TRUE 1
#define FALSE 0
#define MAX_BLOCKS 65000

typedef struct {
    int id;
    int age;
    int height;
} Person;

// Optimize global kernel function
__global__ void analyze_age(Person *people, int *statResults) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    if (id < N) {
        Person person = people[id];

        // Simplify conditional logic
        statResults[id] = (person.age != 28);
    }
}