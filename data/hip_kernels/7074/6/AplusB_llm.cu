#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to add a and b with thread index offset
__global__ void AplusB(int *ret, int a, int b) {
    // Using blockIdx.x and blockDim.x to support multiple blocks
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    ret[index] = a + b + index;
}