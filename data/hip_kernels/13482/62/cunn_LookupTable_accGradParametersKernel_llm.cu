#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_LookupTable_accGradParametersKernel( float *input, float *indices, float *gradOutput, float *gradWeight, float *count, float defaultScale, long numel, long stride, int paddingValue) {

    int idx = blockIdx.x * 4 + threadIdx.y;

    // Number of values processed by each thread (grain size)
    const int SZ = 4;

    if (idx < numel
    && (idx == 0 || input[idx] != input[idx - 1])
    && input[idx] != paddingValue) {
        do {
            const int startFeature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
            const int weightRow = ((int) input[idx] - 1) * stride;
            const int gradOutputRow = ((int) indices[idx] - 1) * stride;
            const float scale = count ? defaultScale / count[idx] : defaultScale;

            float gradient[SZ] = {0};  // Initialize gradient array
            float weight[SZ] = {0};    // Initialize weight array

            #pragma unroll
            for (int ii = 0; ii < SZ; ii++)
            {
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride)
                {
                    gradient[ii] = gradOutput[gradOutputRow + featureDim];
                    weight[ii] = gradWeight[weightRow + featureDim];
                }
            }

            #pragma unroll
            for (int ii = 0; ii < SZ; ii++)
            {
                weight[ii] += gradient[ii] * scale;
            }

            #pragma unroll
            for (int ii = 0; ii < SZ; ii++)
            {
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride)
                {
                    gradWeight[weightRow + featureDim] = weight[ii];
                }
            }

            idx++;
        } while (idx < numel && input[idx] == input[idx - 1]);
    }
}