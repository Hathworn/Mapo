#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomicadaptivemaxgradinput( float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w )
{
    // compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx_start = threadIdx.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    
    int xx_step = blockDim.x;
    int yy_step = blockDim.y * gridDim.y;

    // select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // compute gradInput
    for(int yy = yy_start; yy < output_h; yy += yy_step) {
        int y_start = static_cast<int>(floorf(static_cast<float>(yy) / output_h * input_h));  // Optimize division with floor and cast
        for(int xx = xx_start; xx < output_w; xx += xx_step) {
            int x_start = static_cast<int>(floorf(static_cast<float>(xx) / output_w * input_w));  // Optimize division with floor and cast

            float* ptr_gradInput = gradInput + y_start * input_w + x_start;
            float* ptr_gradOutput = gradOutput + yy * output_w + xx;
            float* ptr_ind_x = indices_x + yy * output_w + xx;
            float* ptr_ind_y = indices_y + yy * output_w + xx;

            float z = *ptr_gradOutput;

            int argmax_x = static_cast<int>(*ptr_ind_x) - 1;  // Explicit type conversion for clarity
            int argmax_y = static_cast<int>(*ptr_ind_y) - 1;  // Explicit type conversion for clarity

            // atomic add since different threads could update same variable
            atomicAdd(&(ptr_gradInput[argmax_x + argmax_y * input_w]), z);
        }
    }
}