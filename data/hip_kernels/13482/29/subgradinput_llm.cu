#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinput(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Compute output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute indices based on block/thread ID
    int o = blockIdx.x;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y * gridDim.y;

    // Adjust pointers for specific output/input planes
    gradOutput += o * output_w * output_h;
    gradInput += o * input_w * input_h;
    
    // Load weight into shared memory (if needed)
    float the_weight = weight[k];

    // Iterate over the output height and width
    for (int yy = yy_start; yy < yy_end; yy += yy_step) {
        for (int xx = xx_start; xx < xx_end; xx += xx_step) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput * the_weight;
            
            // Unroll loops to minimize overhead
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    ptr_gradInput[kx] += z;
                }
                ptr_gradInput += input_w;
            }
        }
    }
}