#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate global thread index
    int global_thread_idx = threadIdx.x + blockIdx.y * blockDim.x;
    
    // Check if thread is within bounds
    if (global_thread_idx < output_w) {
        // Calculate base addresses
        int batch_offset = blockIdx.x * input_w * input_n;
        int thread_offset_input = global_thread_idx * input_n * dW;
        int thread_offset_output = global_thread_idx * input_n;

        float *gradInput_data = gradInput + batch_offset + thread_offset_input;
        float *gradOutput_data = gradOutput + batch_offset + thread_offset_output;
        float *indices_data = indices + batch_offset + thread_offset_output;

        // Update gradInput using feature index from indices
        for (int feat = 0; feat < input_n; ++feat) {
            int index = static_cast<int>(indices_data[feat]);
            atomicAdd(&gradInput_data[index * input_n + feat], gradOutput_data[feat]);
        }
    }
}