#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii/d3;
    z = ii % d2;
    ii = ii/d2;
    y = ii % d1;
    ii = ii/d1;
    x = ii;
    w = w/scale_factor;
    z = z/scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x*d1+y)*d2)+z)*d3+w;
}

__global__ void upscale(float *input, float *output, long no_elements, int scale_factor, int d1, int d2, int d3) {
    // Calculate global index directly using blockIdx and blockDim
    long ii = blockIdx.x * blockDim.x + threadIdx.x;
    ii += (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x;
    
    // Exit if out of bounds
    if (ii >= no_elements) return;
    
    // Translate input index using the device function
    int ipidx = translate_idx(ii, d1, d2, d3, scale_factor);
    
    // Write to output directly from input using translated index
    output[ii] = input[ipidx];
}