#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // Compute global thread ID for efficient memory access
    int xx = threadIdx.x + blockIdx.x * blockDim.x;
    int yy = threadIdx.y + blockIdx.y * blockDim.y;

    if (xx < output_w && yy < output_h) // Boundary check
    {
        int o = blockIdx.z; // Use blockIdx.z for handling batches/input_n

        // Offsets for current output plane
        gradOutput += o * output_w * output_h;
        gradInput += o * input_w * input_h;
        indices_x += o * output_w * output_h;
        indices_y += o * output_w * output_h;

        // Calculate input coordinates
        int y_start = (int)floorf(float(yy) / output_h * input_h);
        int x_start = (int)floorf(float(xx) / output_w * input_w);

        float *ptr_gradInput = gradInput + y_start * input_w + x_start;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float *ptr_ind_x = indices_x + yy * output_w + xx;
        float *ptr_ind_y = indices_y + yy * output_w + xx;
        float z = *ptr_gradOutput;

        int argmax_x = (*ptr_ind_x) - 1;
        int argmax_y = (*ptr_ind_y) - 1;

        atomicAdd(&ptr_gradInput[argmax_x + argmax_y * input_w], z); // Atomic operation for thread safety
    }
}