#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int k = blockIdx.x % input_n;
    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;
    
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    output += o * output_w * output_h;
    input += o * input_w * input_h;

    // Get weight and bias for current output
    float the_weight = weight[k];
    float the_bias = bias[k];

    // Iterating over output pixels
    for (int yy = yy_start; yy < yy_end; yy += yy_step) {
        for (int xx = xx_start; xx < xx_end; xx += xx_step) {
            // Compute the mean of the input image
            float sum = 0.0f;
            float *ptr_input = input + yy * dH * input_w + xx * dW;

            #pragma unroll // Unroll inner loop for better performance
            for (int ky = 0; ky < kH; ky++) {
                #pragma unroll // Unroll inner loop for better performance
                for (int kx = 0; kx < kW; kx++) {
                    sum += ptr_input[kx];
                }
                ptr_input += input_w; // Next input line
            }

            // Update output
            output[yy * output_w + xx] = the_weight * sum + the_bias;
        }
    }
}