#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate the global index for gradInput_data, gradOutput_data, and indices_data
    int batchIndex = blockIdx.x;
    int timeIndex = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    
    if (timeIndex < output_w) {
        float *gradInput_data = gradInput + batchIndex * input_w * input_n + timeIndex * input_n * dW;
        float *gradOutput_data = gradOutput + batchIndex * output_w * input_n + timeIndex * input_n;
        float *indices_data = indices + batchIndex * output_w * input_n + timeIndex * input_n;

        // Loop over all features
        for (int feat = 0; feat < input_n; ++feat) {
            int index = (int)indices_data[feat] * input_n + feat;
            atomicAdd(&gradInput_data[index], gradOutput_data[feat]); // Atomic addition to prevent race condition
        }
    }
}