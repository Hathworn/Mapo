#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void im2col_kernel(int n, float* data_im, int height, int width, int ksize_h, int ksize_w, int pad_h, int pad_w, int stride_h, int stride_w, int dilation_h, int dilation_w, int height_col, int width_col, float* data_col) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;
    while (index < n) {
        // Calculate output dimensions
        int w_out = index % width_col;
        int h_out = (index / width_col) % height_col;
        int channel_in = index / (width_col * height_col);
        int channel_out = channel_in * ksize_h * ksize_w;
        
        // Calculate input dimensions
        int h_in = h_out * stride_h - pad_h;
        int w_in = w_out * stride_w - pad_w;
        
        // Initialize pointers
        float* data_col_ptr = data_col + (channel_out * height_col + h_out) * width_col + w_out;
        float* data_im_ptr = data_im + (channel_in * height + h_in) * width + w_in;
        
        // Iterate over the kernel size
        for (int i = 0; i < ksize_h; ++i) {
            for (int j = 0; j < ksize_w; ++j) {
                // Calculate input position
                int h = h_in + i * dilation_h;
                int w = w_in + j * dilation_w;
                
                // Conditional assignment to avoid out-of-bound access
                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                                data_im_ptr[i * dilation_h * width + j * dilation_w] : 0;
                
                // Move to the next position in the output
                data_col_ptr += height_col * width_col;
            }
        }
        
        // Move to the next index handled by the thread
        index += total_threads;
    }
}