#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void encode_cols_kernel(float *a, uint32_t* b, int m, int n) {
    // Improve thread indexing for better memory coalescing
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = (blockIdx.y * blockDim.y + threadIdx.y) * ENCODE_BITS;
    
    // Process multiple elements per thread where possible
    if (j < n && i < m) {
        uint32_t r = 0;
        for (int k = 0; k < ENCODE_BITS && (i + k) < m; k++) {
            r |= (a[j + n * (i + k)] > 0) << k;
        }
        b[j + n * (i / ENCODE_BITS)] = r;  // Write result back to global memory
    }
}