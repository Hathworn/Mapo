#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binary_gemm_kernel(uint32_t* A, uint32_t* B, float* C, int m, int nn, int k, int transb, int alpha, int beta, float *alphas) {
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    int row = threadIdx.y;
    int col = threadIdx.x;

    int n = 1 + (nn-1)/ENCODE_BITS;
    int startLocation = BLOCK_SIZE * k * blockRow + BLOCK_SIZE * blockCol;

    float* Csub = &C[BLOCK_SIZE * k * blockRow + BLOCK_SIZE * blockCol];

    __shared__ uint32_t As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ uint32_t Bs[BLOCK_SIZE][BLOCK_SIZE];

    int Cvalue = 0;

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int lim = 1 + (n-1) / BLOCK_SIZE;

    // Unroll outer loop to exploit spatial locality
    #pragma unroll
    for (int i = 0; i < lim; ++i) {
        // Get sub-matrix Asub of A
        uint32_t* Asub = &A[BLOCK_SIZE * blockRow * n + BLOCK_SIZE * i];

        // Get sub-matrix Bsub of B
        uint32_t* Bsub = transb ? &B[BLOCK_SIZE * blockCol * n + BLOCK_SIZE * i] : &B[BLOCK_SIZE * k * i + BLOCK_SIZE * blockCol];

        // Load data into shared memory with boundary checks
        if ((BLOCK_SIZE * i + col) < n && r < m)
            As[row][col] = Asub[row * n + col];
        else
            As[row][col] = 0;
        
        if ((BLOCK_SIZE * i + row) < n && c < k)
            Bs[row][col] = transb ? Bsub[row + col * n] : Bsub[row * k + col];
        else
            Bs[row][col] = 0;

        __syncthreads();

        // Inner product with prefetch for data reuse
        #pragma unroll
        for (int j = 0; j < BLOCK_SIZE; ++j)
            Cvalue += __popc(As[row][j] ^ Bs[j][col]);

        __syncthreads();
    }

    // Write to C with boundary checks and adjust if necessary
    if (col + blockCol * BLOCK_SIZE < k && row + blockRow * BLOCK_SIZE < m) {
        float beta_val = beta ? Csub[row * k + col] : 0.0f;
        float adjusted_alphas = alpha ? alphas[(startLocation + row * k + col) / k] : 1.0f;
        Csub[row * k + col] = beta_val + (1.0f * nn - (Cvalue << 1)) * adjusted_alphas;
    }
}