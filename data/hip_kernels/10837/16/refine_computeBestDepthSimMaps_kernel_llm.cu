#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float refineDepthSubPixel(const float3& depths, const float3& sims)
{
    float outDepth = -1.0f;

    float simM1 = (sims.x + 1.0f) / 2.0f; // Normalize simM1
    float simP1 = (sims.z + 1.0f) / 2.0f; // Normalize simP1
    float sim1 = (sims.y + 1.0f) / 2.0f;  // Normalize sim1

    if ((simM1 > sim1) && (simP1 > sim1))
    {
        float dispStep = -((simP1 - simM1) / (2.0f * (simP1 + simM1 - 2.0f * sim1)));

        float floatDepthM1 = depths.x;
        float floatDepthP1 = depths.z;

        float b = (floatDepthP1 + floatDepthM1) / 2.0f; // Calculate linear function intercept
        float a = b - floatDepthM1;                      // Calculate slope

        outDepth = a * dispStep + b;                     // Compute refined depth
    };

    return outDepth;
}

__global__ void refine_computeBestDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float3* isims, int isims_p, float3* idpts, int idpts_p, int width, int height, float simThr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if ((x < width) && (y < height))
    {
        float3 depths = idpts[y * idpts_p + x];
        float3 sims = isims[y * isims_p + x];

        // Choose the depth and similarity with minimum cost.
        float outDepth = ((sims.x < sims.y) ? depths.x : depths.y);
        float outSim = ((sims.x < sims.y) ? sims.x : sims.y);
        outDepth = ((sims.z < outSim) ? depths.z : outDepth);
        outSim = ((sims.z < outSim) ? sims.z : outSim);

        float refinedDepth = refineDepthSubPixel(depths, sims);
        if (refinedDepth > 0.0f)
        {
            outDepth = refinedDepth; // Apply refined depth if valid
        };

        osim[y * osim_p + x] = (outSim < simThr ? outSim : 1.0f); // Apply similarity threshold
        odpt[y * odpt_p + x] = (outSim < simThr ? outDepth : -1.0f); // Apply depth threshold
    };
}