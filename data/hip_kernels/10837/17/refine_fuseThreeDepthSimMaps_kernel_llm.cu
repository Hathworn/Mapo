#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_fuseThreeDepthSimMaps_kernel(float* osim, int osim_p, float* odpt, int odpt_p, float* isimLst, int isimLst_p, float* idptLst, int idptLst_p, float* isimAct, int isimAct_p, float* idptAct, int idptAct_p, int width, int height, float simThr)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure x and y are within bounds
    if(x >= width || y >= height) return;

    // Load data into local variables
    float dpts[3] = {
        odpt[y * odpt_p + x],
        idptLst[y * idptLst_p + x],
        idptAct[y * idptAct_p + x]
    };
    float sims[3] = {
        osim[y * osim_p + x],
        isimLst[y * isimLst_p + x],
        isimAct[y * isimAct_p + x]
    };

    // Compute output depth and similarity values
    float outDpt = dpts[0];
    float outSim = simThr;
    if(sims[1] < outSim) {
        outDpt = dpts[1];
        outSim = sims[1];
    }
    if(sims[2] < outSim) {
        outDpt = dpts[2];
        outSim = sims[2];
    }

    // Store the results
    osim[y * osim_p + x] = outSim;
    odpt[y * odpt_p + x] = outDpt;
}