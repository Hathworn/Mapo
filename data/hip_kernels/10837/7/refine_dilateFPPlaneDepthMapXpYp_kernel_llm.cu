#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refine_dilateFPPlaneDepthMapXpYp_kernel(float* fpPlaneDepthMap, int fpPlaneDepthMap_p, float* maskMap, int maskMap_p, int width, int height, int xp, int yp, float fpPlaneDepth)
{
    // Calculate global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Pre-calculate offsets
    int x_offset = x + xp;
    int y_offset = y + yp;

    // Check boundaries and perform computation
    if(x_offset >= 0 && y_offset >= 0 && x_offset < width && y_offset < height && x < width && y < height)
    {
        float depth = maskMap[y * maskMap_p + x];

        if(depth > 0.0f)
        {
            fpPlaneDepthMap[y_offset * fpPlaneDepthMap_p + x_offset] = fpPlaneDepth;
        }
    }
}