#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generateGaussian_kernel(float* __restrict__ og, float delta, int radius)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x - radius; // Use global indexing to reduce bank conflicts
    float normalization = 1.0f / (2 * delta * delta); // Pre-compute constant expression outside the loop
    og[threadIdx.x] = __expf(-((float)(x * x)) * normalization); // Reuse pre-computed value, cast x*x to float
}