#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDRectifier_backPropagate_kernel(double* x, double* dx, unsigned int size, double leakSlope, int shifting, double clipping)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Precompute shifting factors outside the loop to avoid redundant calculations
    const double shiftFactor = (shifting > 0) ? (1.0 / (1 << shifting)) : (1 << (-shifting));

    for (unsigned int i = index; i < size; i += stride) {
        dx[i] *= shiftFactor;  // Apply shifting factor

        // Use a single conditional statement for clarity and efficiency
        if (clipping > 0.0) {
            dx[i] *= (x[i] > clipping) ? 0.0 : ((x[i] > 0.0) ? 1.0 : leakSlope);
        }
        else {
            dx[i] *= (x[i] > 0.0) ? 1.0 : leakSlope;
        }
    }
}