#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaUpdateFiringRate_kernel(unsigned int *firingRate, unsigned int *totalFiringRatePartial, unsigned int inputsDimX, unsigned int inputsDimY, unsigned int inputsDimZ)
{
    const unsigned int inputSize = inputsDimZ * inputsDimX * inputsDimY;
    const unsigned int batchInputOffset = blockIdx.z * inputSize;
    const unsigned int blockOffset = blockIdx.x * blockDim.x;
    const unsigned int partialIdx = threadIdx.x + blockOffset;

    extern __shared__ unsigned int partialSum[];

    // Ensure partialSum is initialized with zero if partialIdx out of bounds
    if (partialIdx < inputSize) {
        partialSum[threadIdx.x] = firingRate[partialIdx + batchInputOffset];
    } else {
        partialSum[threadIdx.x] = 0;
    }

    __syncthreads();

    // Optimized reduction
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            partialSum[threadIdx.x] += partialSum[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Write final result to global memory
    if (threadIdx.x == 0) {
        totalFiringRatePartial[blockIdx.x + gridDim.x * blockIdx.z] = partialSum[0];
    }
}