#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSReduceIndex_kernel(const unsigned int inputSize, const unsigned int inputBatchOffset, const unsigned int outputBatchOffset, const float* __restrict__ valueThreshold, const float* __restrict__ inputs, int* __restrict__ outputMap, float* __restrict__ scores)
{
    const int batchPos = blockIdx.z;
    const int clsPos = blockIdx.y;
    const int index = (threadIdx.x & 0x1f) + blockIdx.x * blockDim.x;

    // Calculate indices once, reuse in both branches
    const int inputIndex = index + inputSize * clsPos + batchPos * inputBatchOffset;
    const int outputIndex = index + inputSize * clsPos + batchPos * outputBatchOffset;

    if(index < inputSize)
    {
        float value = inputs[inputIndex];

        // Simplify branch with ternary operator
        outputMap[outputIndex] = (value >= valueThreshold[clsPos]) ? index : -1;
        scores[outputIndex] = (value >= valueThreshold[clsPos]) ? value : -1.0;
    }
}