#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSBilinearTF_BackWard_kernel(unsigned int outputWidth, unsigned int outputHeight, unsigned int nbChannels, unsigned int batchSize, unsigned int inputWidth, unsigned int inputHeight, const float scaleX, const float scaleY, const float* diffInput, float* diffOutputs)
{
    // Calculate input and output offsets based on batch and channel indices
    const unsigned int inputOffset = (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels * inputWidth * inputHeight;
    const unsigned int outputOffset = (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels * outputWidth * outputHeight;

    for (unsigned int ch = blockIdx.x; ch < nbChannels; ch += gridDim.x) {
        for (unsigned int oy = threadIdx.y; oy < outputHeight; oy += blockDim.y) {
            // Precompute Y coordinate information
            const float in_y = oy * scaleY;
            const int top_y_index = (int)(floorf(in_y));
            const int bottom_y_index = (in_y < inputHeight - 1) ? ceilf(in_y) : inputHeight - 1;
            const float y_lerp = in_y - top_y_index;
            const float inverse_y_lerp = 1.0f - y_lerp;

            for (unsigned int ox = threadIdx.x; ox < outputWidth; ox += blockDim.x) {
                // Precompute X coordinate information
                const float in_x = ox * scaleX;
                const int left_x_index = (int)(floorf(in_x));
                const int right_x_index = (in_x < inputWidth - 1) ? ceilf(in_x) : inputWidth - 1;
                const float x_lerp = in_x - left_x_index;
                const float inverse_x_lerp = 1.0f - x_lerp;

                // Calculate indices in the input and output arrays
                const unsigned int inLeftTopIdx = left_x_index + top_y_index * inputWidth + ch * inputWidth * inputHeight + inputOffset;
                const unsigned int inRightTopIdx = right_x_index + top_y_index * inputWidth + ch * inputWidth * inputHeight + inputOffset;
                const unsigned int inLeftBotIdx = left_x_index + bottom_y_index * inputWidth + ch * inputWidth * inputHeight + inputOffset;
                const unsigned int inRightBotIdx = right_x_index + bottom_y_index * inputWidth + ch * inputWidth * inputHeight + inputOffset;

                const unsigned int outIdx = ox + oy * outputWidth + ch * outputWidth * outputHeight + outputOffset;
                const float outData = diffInput[outIdx];

                // Use atomic operations to avoid race conditions
                atomicAdd(&diffOutputs[inLeftTopIdx], outData * inverse_y_lerp * inverse_x_lerp);
                atomicAdd(&diffOutputs[inRightTopIdx], outData * inverse_y_lerp * x_lerp);
                atomicAdd(&diffOutputs[inLeftBotIdx], outData * y_lerp * inverse_x_lerp);
                atomicAdd(&diffOutputs[inRightBotIdx], outData * y_lerp * x_lerp);
            }
        }
    }
}