#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSetupRng_kernel(hiprandState * state, unsigned int seed)
{
    // Calculate the global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialize RNG with unique sequence for each thread
    hiprand_init(seed, id, 0, &state[id]);
}