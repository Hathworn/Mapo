#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSBilinearTF_Forward_kernel(unsigned int outputWidth, unsigned int outputHeight, unsigned int nbChannels, unsigned int batchSize, unsigned int inputWidth, unsigned int inputHeight, const unsigned int* yLowIdx, const unsigned int* yHighIdx, const float* yInter, const unsigned int* xLowIdx, const unsigned int* xHighIdx, const float* xInter, const float* input, float* outputs)
{
    const unsigned int inputOffset = (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels * inputWidth * inputHeight;
    const unsigned int outputOffset = (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels * outputWidth * outputHeight;

    unsigned int ch = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int oy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ch < nbChannels && oy < outputHeight) {
        for (unsigned int ox = threadIdx.x; ox < outputWidth; ox += blockDim.x) { // Loop x dimension

            // Calculate input indices
            const unsigned int indexTL = xLowIdx[ox] + yLowIdx[oy] * inputWidth + ch * inputWidth * inputHeight + inputOffset;
            const unsigned int indexTR = xHighIdx[ox] + yLowIdx[oy] * inputWidth + ch * inputWidth * inputHeight + inputOffset;
            const unsigned int indexBL = xLowIdx[ox] + yHighIdx[oy] * inputWidth + ch * inputWidth * inputHeight + inputOffset;
            const unsigned int indexBR = xHighIdx[ox] + yHighIdx[oy] * inputWidth + ch * inputWidth * inputHeight + inputOffset;

            // Fetch input values
            const float top_left = input[indexTL];
            const float top_right = input[indexTR];
            const float bottom_left = input[indexBL];
            const float bottom_right = input[indexBR];

            // Perform bilinear interpolation
            const float top = top_left + (top_right - top_left) * xInter[ox];
            const float bottom = bottom_left + (bottom_right - bottom_left) * xInter[ox];

            // Store result in output array
            outputs[ox + oy * outputWidth + ch * outputWidth * outputHeight + outputOffset] = top + (bottom - top) * yInter[oy];
        }
    }
}