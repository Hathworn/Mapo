#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSSaturation_backPropagate_kernel(float* x, float* dx, unsigned int size, int shifting, float threshold)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Loop unrolling for potential performance improvement
    for (unsigned int i = index; i < size; i += stride * 4) {
        #pragma unroll // Unrolling the loop for 4 iterations
        for (unsigned int j = 0; j < 4 && i + j < size; ++j) {
            unsigned int idx = i + j;
            if (shifting > 0)
                dx[idx] /= (1 << shifting);
            else if (shifting < 0)
                dx[idx] *= (1 << (-shifting));

            if (threshold != 0.0f) {
                dx[idx] *= (x[idx] > -threshold && x[idx] < threshold) ? 1.0f : 0.0f;
            }
        }
    }
}