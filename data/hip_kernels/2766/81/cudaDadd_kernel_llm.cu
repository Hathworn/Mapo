#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDadd_kernel(unsigned int size, double value, const double *x, double *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        y[index] = x[index] + value;  // Compute directly on element if within bounds
    }
}