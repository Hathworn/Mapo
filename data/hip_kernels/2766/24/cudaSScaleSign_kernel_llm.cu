#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSScaleSign_kernel(unsigned int size, float* input, float* sign, const float scale, const float beta, float* result)
{
    // Determine global thread index
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory to reduce redundant computation
    __shared__ float shared_scale;
    __shared__ float shared_beta;
    if(threadIdx.x == 0) {
        shared_scale = scale;
        shared_beta = beta;
    }
    __syncthreads();

    // Kernel operation with conditional logic outside loop
    for (unsigned int i = index; i < size; i += stride) {
        const float sgn = (sign[i] >= 0) ? 1.0f : -1.0f;
        float scaled_input = input[i] * sgn * shared_scale; // Precompute scaled input
        
        // Simplify conditional within loop
        if (shared_beta != 0.0f) {
            result[i] = scaled_input + shared_beta * result[i];
        } else {
            result[i] = scaled_input;
        }
    }
}