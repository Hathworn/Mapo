#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSPadding_kernel(unsigned int nbOutputs, unsigned int outputWidth, unsigned int outputHeight, unsigned int nbChannels, unsigned int batchSize, unsigned int inputWidth, unsigned int inputHeight, int leftPad, int rightPad, int topPad, int botPad, const float* input, float* outputs) {

    const unsigned int inputOffset = (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels * inputWidth * inputHeight;
    const unsigned int outputOffset = (blockIdx.z * blockDim.z + threadIdx.z) * nbOutputs * outputWidth * outputHeight;

    // nbCh = nbChannels for propagate = nbOutputs for back-propagate
    const unsigned int nbCh = min(nbChannels, nbOutputs);

    for (unsigned int ch = blockIdx.x; ch < nbCh; ch += gridDim.x) {
        for (unsigned int oy = threadIdx.y; oy < outputHeight; oy += blockDim.y) {
            for (unsigned int ox = threadIdx.x; ox < outputWidth; ox += blockDim.x) {

                int ix = (int)ox - leftPad;
                int iy = (int)oy - topPad;

                // Use shared memory for faster access
                __shared__ float sharedInput[blockDim.y][blockDim.x];
                float outputValue = 0.0f;

                if (ix >= 0 && ix < (int)inputWidth && iy >= 0 && iy < (int)inputHeight) {
                    // Load from input into shared memory
                    sharedInput[threadIdx.y][threadIdx.x] = input[ix + iy * inputWidth + ch * inputWidth * inputHeight + inputOffset];
                    __syncthreads();
                    outputValue = sharedInput[threadIdx.y][threadIdx.x];
                }

                outputs[ox + oy * outputWidth + ch * outputWidth * outputHeight + outputOffset] = outputValue;
            }
        }
    }
}