#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSclamp_kernel(float* x, unsigned int size, float minVal, float maxVal)
{
    // Calculate the index for the current thread
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Iterate with stride, process elements within bounds
    for (unsigned int i = index; i < size; i += stride) {
        float val = x[i];
        // Use fminf and fmaxf for potential performance gain
        x[i] = fmaxf(minVal, fminf(val, maxVal));
    }
}