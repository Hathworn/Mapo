#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSinv_kernel(unsigned int size, const float *x, float *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Optimize memory access by using shared memory
    __shared__ float sharedX[blockDim.x]; 

    for (unsigned int i = index; i < size; i += stride) {
        // Load data into shared memory
        sharedX[threadIdx.x] = x[i];

        __syncthreads(); // Ensure all threads have loaded their data

        // Compute reciprocal using shared memory
        y[i] = 1.0f / sharedX[threadIdx.x];

        __syncthreads(); // Ensure computation is finished before next load
    }
}