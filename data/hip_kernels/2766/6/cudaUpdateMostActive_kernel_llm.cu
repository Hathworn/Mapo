#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaUpdateMostActive_kernel(unsigned int * exampleIds, unsigned int * exampleFiringRate, unsigned int * mostActiveId, unsigned int inputsDimX, unsigned int inputsDimY, unsigned int inputsDimZ)
{
    const unsigned int inputSize = inputsDimZ * inputsDimX * inputsDimY;
    const unsigned int batchInputOffset = blockIdx.z * inputSize;
    const unsigned int blockOffset = blockIdx.x * blockDim.x;
    const unsigned int partialIdx = threadIdx.x + blockOffset;

    extern __shared__ unsigned int partialActiveIdx[];
    extern __shared__ unsigned int partialFiringRate[];

    // Initialize shared memory with firing rates
    if (partialIdx < inputSize) {
        partialActiveIdx[threadIdx.x] = exampleIds[partialIdx + batchInputOffset];
        partialFiringRate[threadIdx.x] = exampleFiringRate[partialActiveIdx[threadIdx.x]];
    } else {
        partialFiringRate[threadIdx.x] = 0;
        partialActiveIdx[threadIdx.x] = 0;
    }

    __syncthreads();

    // Reduction over neurons
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            if (partialFiringRate[threadIdx.x] < partialFiringRate[threadIdx.x + offset]) {
                partialFiringRate[threadIdx.x] = partialFiringRate[threadIdx.x + offset];
                partialActiveIdx[threadIdx.x] = partialActiveIdx[threadIdx.x + offset];
            }
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        mostActiveId[blockIdx.x + gridDim.x * blockIdx.z] = partialActiveIdx[0];
    }
}