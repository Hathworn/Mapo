#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSSaturation_propagate_kernel(float* x, float* y, unsigned int size, int shifting, float threshold)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory to avoid repeated shift calculations when possible
    __shared__ float shiftFactor;
    if (threadIdx.x == 0) {
        shiftFactor = (shifting > 0) ? (1.0f / (1 << shifting)) : ((shifting < 0) ? (1 << (-shifting)) : 1.0f);
    }
    __syncthreads();

    for (unsigned int i = index; i < size; i += stride) {
        float value = x[i] * shiftFactor; // Apply the precomputed shift factor

        // Saturate 'value' using threshold
        if (threshold != 0.0f) {
            y[i] = (value < -threshold) ? -threshold
                 : (value > threshold) ? threshold
                 : value;
        } else {
            y[i] = value; // Direct copy if no threshold
        }
    }
}