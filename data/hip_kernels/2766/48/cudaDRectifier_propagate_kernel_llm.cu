#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDRectifier_propagate_kernel(double* x, double* y, unsigned int size, double leakSlope, int shifting, double clipping)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory to reduce redundant global memory access
    __shared__ double shared_clipping;
    __shared__ int shared_shifting;
    __shared__ double shared_leakSlope;

    if (threadIdx.x == 0) {
        shared_clipping = clipping;
        shared_shifting = shifting;
        shared_leakSlope = leakSlope;
    }
    __syncthreads();

    for (unsigned int i = index; i < size; i += stride) {
        double value = x[i];

        // Precompute shifting effect
        if (shared_shifting != 0) {
            value = shared_shifting > 0 ? value / (1 << shared_shifting) : value * (1 << (-shared_shifting));
        }

        // Optimize conditional assignment using shared memory
        if (shared_clipping > 0.0) {
            y[i] = (value > 0.0) ? min(value, shared_clipping) : shared_leakSlope * value;
        } else {
            y[i] = (value > 0.0) ? value : shared_leakSlope * value;
        }
    }
}