#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to accumulate values from y to x
__global__ void cudaDaccumulate_kernel(double* x, int* y, unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Using index within bounds to prevent out-of-bounds access
    for (unsigned int i = index; i < size; i += stride) {
        x[i] += __ldg(&y[i]);  // Use __ldg() for potential cache benefit
    }
}