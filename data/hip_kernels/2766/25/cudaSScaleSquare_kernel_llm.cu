#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSScaleSquare_kernel(unsigned int size, float* input, const float scale, const float shift, const float beta, float* result)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Optimize by reducing branching
    float betaFactor = (beta != 0.0f) ? beta : 0.0f;

    for (unsigned int i = index; i < size; i += stride) {
        float val = input[i] * input[i] * scale + shift;
        result[i] = (beta != 0.0f) ? (val + beta * result[i]) : val;
    }
}