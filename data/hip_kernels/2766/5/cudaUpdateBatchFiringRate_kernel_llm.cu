#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaUpdateBatchFiringRate_kernel(unsigned int * firingRate, unsigned int * batchFiringRate, unsigned int inputsDimX, unsigned int inputsDimY, unsigned int inputsDimZ, unsigned int batchSize)
{
    const unsigned int inputSize = inputsDimZ * inputsDimX * inputsDimY;
    const unsigned int channel = blockIdx.x;
    const unsigned int sy = threadIdx.y;
    const unsigned int sx = threadIdx.x;

    if (channel < inputsDimZ && sy < inputsDimY && sx < inputsDimX) {
        // Compute the input index for the current thread
        const unsigned int inputsIdx = channel * inputsDimX * inputsDimY + sy * inputsDimX + sx;

        unsigned int batchSum = 0;
        for (unsigned int batch = 0; batch < batchSize; ++batch) {
            const unsigned int batchInputOffset = batch * inputSize;
            batchSum += firingRate[inputsIdx + batchInputOffset];
        }
        batchFiringRate[inputsIdx] = batchSum;
    }
}