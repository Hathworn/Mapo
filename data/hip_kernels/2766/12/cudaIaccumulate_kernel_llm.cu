#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaIaccumulate_kernel(int* x, int* y, unsigned int size)
{
    // Using more variables in registers for optimization
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    int lx, ly;

    for (unsigned int i = index; i < size; i += stride) {
        // Load x[i] and y[i] to local variables
        lx = x[i];
        ly = y[i];
        // Perform the addition in registers
        lx += ly;
        // Store the result back to global memory
        x[i] = lx;
    }
}