#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSmult_kernel(unsigned int size, const float *x1, const float *x2, float *y)
{
    // Calculate global thread index
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use consistency with built-in variable names
    if (index < size) {
        // Perform the multiplication for the current index
        y[index] = x1[index] * x2[index];
    }
}