#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSquantize_kernel(float* x, float* y, unsigned int size, float minVal, float maxVal, unsigned int quantizationLevels, bool truncate)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    if (quantizationLevels > 1) {
        const float scaling = (maxVal - minVal) / (float)(quantizationLevels - 1);

        // Unrolling the loop for better performance
        for (unsigned int i = index; i < size; i += stride*4) {
            float clamped, value;

            #pragma unroll
            for (int j = 0; j < 4; ++j) {
                unsigned int idx = i + j * stride;
                if (idx < size) {
                    clamped = fminf(fmaxf(x[idx], minVal), maxVal);
                    if (truncate)
                        value = (int)((clamped - minVal) / scaling) * scaling + minVal;
                    else
                        value = (int)roundf((clamped - minVal) / scaling) * scaling + minVal;

                    y[idx] = value;
                }
            }
        }
    }
    else {
        // Unrolling the loop for better performance
        for (unsigned int i = index; i < size; i += stride*4) {
            #pragma unroll
            for (int j = 0; j < 4; ++j) {
                unsigned int idx = i + j * stride;
                if (idx < size) {
                    y[idx] = ((x[idx] >= 0.0f) ? 1.0f : -1.0f);
                }
            }
        }
    }
}