#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaUpdateMostActive_kernel(unsigned int * exampleFiringRate, unsigned int * mostActiveId, unsigned int inputsDimX, unsigned int inputsDimY, unsigned int inputsDimZ)
{
    const unsigned int inputSize = inputsDimZ * inputsDimX * inputsDimY;
    const unsigned int batchInputOffset = blockIdx.z * inputSize;

    extern __shared__ unsigned int partialActiveIdx[];

    unsigned int maxIdx = threadIdx.x;
    unsigned int maxVal = 0;

    // Initialize shared memory and determine max index and value for each thread
    if (threadIdx.x < inputSize) {
        maxVal = exampleFiringRate[threadIdx.x + batchInputOffset];
        for (unsigned int i = threadIdx.x + blockDim.x; i < inputSize; i += blockDim.x) {
            unsigned int val = exampleFiringRate[i + batchInputOffset];
            if (val > maxVal) {
                maxVal = val;
                maxIdx = i;
            }
        }
    }
    partialActiveIdx[threadIdx.x] = maxIdx;

    __syncthreads();

    // Reduction to find the maximum firing rate index
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            unsigned int idx = partialActiveIdx[threadIdx.x + offset];
            if (exampleFiringRate[idx + batchInputOffset] > exampleFiringRate[partialActiveIdx[threadIdx.x] + batchInputOffset]) {
                partialActiveIdx[threadIdx.x] = idx;
            }
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        mostActiveId[blockIdx.z] = partialActiveIdx[0];
    }
}