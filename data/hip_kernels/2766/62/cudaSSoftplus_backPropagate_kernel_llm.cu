#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSSoftplus_backPropagate_kernel(float* x, float* dx, unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {  // Optimize by checking index boundary
        dx[index] *= (1.0f - expf(-x[index]));  // Use expf for better performance with floats
    }
}