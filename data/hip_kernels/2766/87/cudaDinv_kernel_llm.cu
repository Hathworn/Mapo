#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDinv_kernel(unsigned int size, const double *x, double *y)
{
    // Calculate the global index for each thread
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll the loop for better performance
    if (index < size) {
        y[index] = 1.0 / x[index];
        
        index += blockDim.x * gridDim.x;
        if (index < size) {
            y[index] = 1.0 / x[index];
        }
    }
}