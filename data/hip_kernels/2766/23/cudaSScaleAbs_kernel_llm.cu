#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSScaleAbs_kernel(unsigned int size, float* input, const float scale, const float beta, float* result)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    for (unsigned int i = index; i < size; i += stride) {
        float scaledValue = fabs(input[i]) * scale; // Common calculation factored out
        result[i] = (beta != 0.0f) ? (scaledValue + beta * result[i]) : scaledValue; // Use conditional to reduce code duplication
    }
}