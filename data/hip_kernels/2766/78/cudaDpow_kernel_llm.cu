#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDpow_kernel(unsigned int size, double power, const double *x, double *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use double precision pow function for better accuracy
    for (unsigned int i = index; i < size; i += stride) {
        y[i] = pow(x[i], power);
    }
}