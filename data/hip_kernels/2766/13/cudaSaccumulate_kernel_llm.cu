#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSaccumulate_kernel(float* x, int* y, unsigned int size) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory to reduce global memory accesses
    extern __shared__ int shared_y[];
    if (index < size) {
        shared_y[threadIdx.x] = y[index];
        __syncthreads();

        for (unsigned int i = index; i < size; i += stride) {
            x[i] += shared_y[threadIdx.x]; // Use shared memory value
        }
    }
}