#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDSoftplus_propagate_kernel(double* x, double* y, unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Process multiple elements per thread to improve memory throughput
    for (unsigned int i = index; i < size; i += stride) {
        double xi = x[i];
        y[i] = log1p(exp(xi));  // Use log1p for better numerical stability with small values
    }
}