#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDquantize_kernel(double* x, double* y, unsigned int size, double minVal, double maxVal, unsigned int quantizationLevels, bool truncate)
{
    // Calculate global thread index
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Calculate once outside the loop
    const double scaling = (quantizationLevels > 1) ? (maxVal - minVal) / (double)(quantizationLevels - 1) : 0.0;
    
    for (unsigned int i = index; i < size; i += stride) {
        double clamped = (quantizationLevels > 1) ? 
            (x[i] < minVal ? minVal : (x[i] > maxVal ? maxVal : x[i])) :
            x[i];
        
        if (quantizationLevels > 1) {
            // Choose truncation or rounding based on flag
            y[i] = truncate ? 
                ((int)((clamped - minVal) / scaling) * scaling + minVal) :
                ((int)round((clamped - minVal) / scaling) * scaling + minVal);
        } else {
            y[i] = ((x[i] >= 0.0) ? 1.0 : -1.0);
        }
    }
}
```
