#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDclamp_kernel(double* x, unsigned int size, double minVal, double maxVal)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    for (unsigned int i = index; i < size; i += stride) {
        // Use fmax and fmin for clamping
        x[i] = fmax(fmin(x[i], maxVal), minVal);
    }
}