#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSMaxForward_kernel(unsigned int size, float* input, float* maxVal, const unsigned int idx, unsigned int* argMax)
{
    // Calculate global index
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Loop over elements with stride
    for (unsigned int i = index; i < size; i += stride) {
        // Update maxVal and argMax with atomic operation
        if (input[i] > maxVal[i]) {
            atomicMax(reinterpret_cast<int*>(maxVal + i), __float_as_int(input[i]));
            argMax[i] = idx;
        }
    }
}