#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDSaturation_backPropagate_kernel(double* x, double* dx, unsigned int size, int shifting, double threshold)
{
    // Calculate the flattened global index
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory for performance improvement in case it's being reused
    extern __shared__ double shared_dx[];

    if (index < size) {
        // Precompute shift values to reduce branching
        double shift_value = 1;
        if (shifting > 0) {
            shift_value = 1.0 / (1 << shifting);
        } else if (shifting < 0) {
            shift_value = 1 << (-shifting);
        }

        // Iterate over all elements this thread is responsible for
        for (unsigned int i = index; i < size; i += stride) {
            dx[i] *= shift_value;
            if (threshold != 0.0) {
                dx[i] *= (x[i] > -threshold && x[i] < threshold) ? 1.0 : 0.0;
            }
        }
    }
}