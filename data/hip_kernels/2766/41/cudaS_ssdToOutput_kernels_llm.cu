#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaS_ssdToOutput_kernels(  unsigned int batchSize, unsigned int nbClass, unsigned int nbAnchors, unsigned int channelWidth, unsigned int channelHeight, unsigned int nbProposals, unsigned int* nbValidROIs, unsigned int cls, unsigned int totalParts, unsigned int totalTemplates, unsigned int maxParts, unsigned int maxTemplates, unsigned int cumulParts, unsigned int cumulTemplates, unsigned int nbParts, unsigned int nbTemplates, float xRatio, float yRatio, float xOutputRatio, float yOutputRatio, const float* roi_bbox, const float* roi_anchors, const float* anchors, const float* inputs_parts, const float* inputs_templates, float* outputs)
{
    const int batchPos = blockIdx.z;
    const int proposal = (threadIdx.x & 0x1f) + blockIdx.x*blockDim.x;
    const int ptIdx = blockIdx.y;

    // Precompute commonly used index
    const unsigned int n = proposal + cls*nbProposals + batchPos*nbProposals*nbClass;
    const int baseOutputIdx = n * (6 + maxParts*2 + maxTemplates*3);

    const int nbDetectedObject  = (int) nbValidROIs[batchPos];
    if (proposal < nbProposals) {
        if (proposal < nbDetectedObject) {
            if (ptIdx == 0) {
                // Reusing memory layout calculations
                const int baseRoiIdx = 5 * proposal + batchPos * nbProposals * 5;
                for (int i = 0; i < 5; ++i) {
                    outputs[i + baseOutputIdx] = roi_bbox[i + baseRoiIdx];
                }
                outputs[5 + baseOutputIdx] = (float) cls;
            }

            if (ptIdx < nbParts && totalParts > 0) {
                const unsigned int xa = roi_anchors[0 + 5*proposal + batchPos*nbProposals*5];
                const unsigned int ya = roi_anchors[1 + 5*proposal + batchPos*nbProposals*5];
                const unsigned int k  = roi_anchors[2 + 5*proposal + batchPos*nbProposals*5];

                // Use precomputed size multipliers
                const int partOffset = (k * nbParts * 2 + cumulParts + ptIdx * 2) * channelHeight * channelWidth;
                const int batchOffset = batchPos * channelHeight * channelWidth * nbAnchors * 2 * totalParts;
                
                const int yIdx = xa + ya * channelWidth + partOffset + batchOffset;
                const int xIdx = yIdx + channelHeight * channelWidth;

                const float partY = inputs_parts[yIdx];
                const float partX = inputs_parts[xIdx];

                const int xa0 = (int)(anchors[k * 4] + xa * xRatio);
                const int ya0 = (int)(anchors[k * 4 + 1] + ya * yRatio);
                const int xa1 = (int)(anchors[k * 4 + 2] + xa * xRatio);
                const int ya1 = (int)(anchors[k * 4 + 3] + ya * yRatio);

                const int wa = xa1 - xa0;
                const int ha = ya1 - ya0;

                const float xac = xa0 + wa / 2.0f;
                const float yac = ya0 + ha / 2.0f;
                const float predPartY = (partY * ha + yac) * yOutputRatio;
                const float predPartX = (partX * wa + xac) * xOutputRatio;

                const int partsOffset = ptIdx * 2 + 6;
                outputs[partsOffset + baseOutputIdx] = predPartY;
                outputs[partsOffset + 1 + baseOutputIdx] = predPartX;
            }
            else if (ptIdx < maxParts && totalParts > 0) {
                const int partsOffset = ptIdx * 2 + 6;
                outputs[partsOffset + baseOutputIdx] = 0.0;
                outputs[partsOffset + 1 + baseOutputIdx] = 0.0;
            }

            if (ptIdx < nbTemplates && totalTemplates > 0) {
                const unsigned int xa = roi_anchors[0 + 5*proposal + batchPos*nbProposals*5];
                const unsigned int ya = roi_anchors[1 + 5*proposal + batchPos*nbProposals*5];
                const unsigned int k  = roi_anchors[2 + 5*proposal + batchPos*nbProposals*5];

                const int tempOffset = (k * nbTemplates * 3 + cumulTemplates + ptIdx * 3) * channelHeight * channelWidth;
                const int batchTempOffset = batchPos * channelHeight * channelWidth * nbAnchors * 3 * totalTemplates;

                const int yIdx = xa + ya * channelWidth + tempOffset + batchTempOffset;
                const int xIdx = yIdx + channelHeight * channelWidth;
                const int zIdx = xIdx + channelHeight * channelWidth;

                const float templateY = expf(inputs_templates[yIdx]);
                const float templateX = expf(inputs_templates[xIdx]);
                const float templateZ = expf(inputs_templates[zIdx]);

                const int templatesOffset = ptIdx * 3 + maxParts * 2 + 6;
                outputs[templatesOffset + baseOutputIdx] = templateY;
                outputs[templatesOffset + 1 + baseOutputIdx] = templateX;
                outputs[templatesOffset + 2 + baseOutputIdx] = templateZ;
            }
            else if (ptIdx < maxTemplates && totalTemplates > 0) {
                const int templatesOffset = ptIdx * 3 + maxParts * 2 + 6;
                outputs[templatesOffset + baseOutputIdx] = 0.0;
                outputs[templatesOffset + 1 + baseOutputIdx] = 0.0;
                outputs[templatesOffset + 2 + baseOutputIdx] = 0.0;
            }
        }
        else {
            for (int i = 0; i < 5; ++i) {
                outputs[i + baseOutputIdx] = 0.0;
            }

            if (ptIdx < maxParts && totalParts > 0) {
                const int partsOffset = ptIdx * 2 + 6;
                outputs[partsOffset + baseOutputIdx] = 0.0;
                outputs[partsOffset + 1 + baseOutputIdx] = 0.0;
            }

            if (ptIdx < maxTemplates && totalTemplates > 0) {
                const int templatesOffset = ptIdx * 3 + maxParts * 2 + 6;
                for (int i = 0; i < 3; ++i) {
                    outputs[templatesOffset + i + baseOutputIdx] = 0.0;
                }
            }
        }
    }
}