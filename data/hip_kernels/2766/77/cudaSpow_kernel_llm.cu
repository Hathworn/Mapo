#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSpow_kernel(unsigned int size, float power, const float *x, float *y)
{
    // Calculate global index once
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    // Optimize by unrolling the loop
    for (unsigned int i = index; i < size; i += stride * 4) {
        if (i < size) y[i] = powf(x[i], power);
        if (i + stride < size) y[i + stride] = powf(x[i + stride], power);
        if (i + 2 * stride < size) y[i + 2 * stride] = powf(x[i + 2 * stride], power);
        if (i + 3 * stride < size) y[i + 3 * stride] = powf(x[i + 3 * stride], power);
    }
}