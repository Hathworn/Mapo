#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function with loop unrolling and early exit condition
__global__ void cudaDSaturation_propagate_kernel(double* x, double* y, unsigned int size, int shifting, double threshold)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    if(index >= size) return; // Early exit if index is out of bounds

    double factor = (shifting > 0) ? (1.0 / (1 << shifting)) : (1 << (-shifting));
    double th = fabs(threshold); // Use absolute value for consistency

    for (unsigned int i = index; i < size; i += stride) {
        double value = x[i] * factor; // Apply shifting once
        
        // Apply saturation with ternary operators
        y[i] = (th != 0.0) ? ((value < -th) ? -th : ((value > th) ? th : value)) : value;
    }
}