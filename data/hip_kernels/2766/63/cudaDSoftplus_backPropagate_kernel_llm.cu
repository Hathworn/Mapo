#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDSoftplus_backPropagate_kernel(double* x, double* dx, unsigned int size)
{
    // Optimize index calculation and loop control
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        // Utilizing direct thread access, removing the need for a stride-based loop
        dx[index] *= (1.0 - exp(-x[index]));
    }
}