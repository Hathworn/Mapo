#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSAnchorBackPropagateSSD_NegSamples_kernel(const float* inputCls, float* diffOutputsCls, const float* confSamples, const int* keySamples, const int nbSamples, const int nbPositive, const unsigned int nbAnchors, const unsigned int outputsHeight, const unsigned int outputsWidth, const unsigned int batchSize)
{
    // Calculate global index. Combined block and thread indexing for better performance
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Only process if index is within valid range
    if (index < nbSamples) 
    {
        // Access the key sample directly using index
        const int indexSamples = keySamples[index];
        
        // Compute error once, store in a local variable for reuse
        const float error = inputCls[indexSamples];
        
        // Directly compute output using the error and save the result
        diffOutputsCls[indexSamples] = -error / (nbPositive * batchSize);
    }
}