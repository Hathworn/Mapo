#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSMult_kernel(unsigned int size, float* a, float* b, const float beta, float* result)
{
    // Calculate global thread index
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Reduce branching by using a single loop
    for (unsigned int i = index; i < size; i += stride) {
        float temp = a[i] * b[i];
        // Use a conditional operator for concise beta handling
        result[i] = (beta != 0.0f) ? temp + beta * result[i] : temp;
    }
}