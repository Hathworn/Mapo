#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaUpdateActivity_kernel(int * inputs, char * activity, unsigned int * firingRate, unsigned int * exampleFiringRate, int * totalOutput, unsigned long long int * firstEventTime, unsigned long long int * lastEventTime, unsigned int inputsDimX, unsigned int inputsDimY, unsigned int inputsDimZ, unsigned int long long timestamp)
{
    const unsigned int inputSize = inputsDimZ * inputsDimX * inputsDimY;
    const unsigned int batchInputOffset = blockIdx.z * inputSize;

    unsigned int inputIndex = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (blockIdx.x + gridDim.x * blockIdx.y));
    unsigned int inputStride = blockDim.x * blockDim.y * gridDim.x * gridDim.y;

    for (unsigned int i = inputIndex; i < inputsDimX * inputsDimY * inputsDimZ; i += inputStride) {
        unsigned int channel = i / (inputsDimX * inputsDimY);
        unsigned int yz = i % (inputsDimX * inputsDimY);
        unsigned int y = yz / inputsDimX;
        unsigned int x = yz % inputsDimX;

        if (channel < inputsDimZ) {
            unsigned int inputsIdx = x + y * inputsDimX + channel * inputsDimX * inputsDimY;
            int act = inputs[inputsIdx + batchInputOffset];
            unsigned int actAbs = abs(act);
            char spike = act == 0 ? 0 : act / abs(act);

            activity[inputsIdx + batchInputOffset] = spike;
            firingRate[inputsIdx + batchInputOffset] += actAbs;
            exampleFiringRate[inputsIdx + batchInputOffset] += actAbs;
            totalOutput[inputsIdx + batchInputOffset] += act;
        }
    }
}