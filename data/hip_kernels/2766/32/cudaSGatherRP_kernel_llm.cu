#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSGatherRP_kernel(unsigned int inputSizeX, unsigned int inputSizeY, unsigned int nbAnchors, unsigned int batchSize, const float* inputs, const int* i, const int* j, const int* k, const int* b, const int* mask, float* outputs, int* anchors, unsigned int topN, const unsigned int nbProposals)
{
    // Calculate indices
    const int batchPos = blockIdx.z;
    const int sortOffset = batchPos * topN;
    const int index = (threadIdx.x & 0x1f) + blockIdx.x * blockDim.x;
    if (index >= nbProposals) return; // Return early if out of bounds

    const int totalIndex = index + sortOffset;
    const int batchIndex = index + batchPos * nbProposals;
    const int maskIndex = mask[totalIndex] + sortOffset;

    // Pre-calculate common terms
    const int inputSizeXY = inputSizeX * inputSizeY;
    const int anchorOffset = b[maskIndex] * nbAnchors * inputSizeXY * 6;

    // Compute indices
    unsigned int baseIdx = i[maskIndex] + j[maskIndex] * inputSizeX;
    unsigned int xIdx = baseIdx + (k[maskIndex] + nbAnchors) * inputSizeXY + anchorOffset;
    unsigned int yIdx = baseIdx + (k[maskIndex] + 2 * nbAnchors) * inputSizeXY + anchorOffset;
    unsigned int wIdx = baseIdx + (k[maskIndex] + 3 * nbAnchors) * inputSizeXY + anchorOffset;
    unsigned int hIdx = baseIdx + (k[maskIndex] + 4 * nbAnchors) * inputSizeXY + anchorOffset;

    // Store results
    int offset = batchIndex * 4;
    anchors[offset] = i[maskIndex];
    anchors[offset + 1] = j[maskIndex];
    anchors[offset + 2] = k[maskIndex];
    anchors[offset + 3] = b[maskIndex];

    outputs[offset] = inputs[xIdx];
    outputs[offset + 1] = inputs[yIdx];
    outputs[offset + 2] = inputs[wIdx];
    outputs[offset + 3] = inputs[hIdx];
}