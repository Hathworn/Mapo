#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSMaxBackward_kernel(unsigned int size, float* diffInput, const unsigned int idx, unsigned int* argMax, const float beta, float* result)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory for better global memory access pattern
    __shared__ float sharedDiffInput[1024]; // Assuming blockDim.x <= 1024
    __shared__ unsigned int sharedArgMax[1024];

    // Load data into shared memory
    if (index < size) {
        sharedDiffInput[threadIdx.x] = diffInput[index];
        sharedArgMax[threadIdx.x] = argMax[index];
    }
    __syncthreads();

    if (beta != 0.0f) {
        for (unsigned int i = index; i < size; i += stride) {
            // Leveraging shared memory to reduce memory accesses
            result[i] = (sharedArgMax[threadIdx.x] == idx) ? (sharedDiffInput[threadIdx.x] + beta * result[i])
                                                           : beta * result[i];
        }
    } else {
        for (unsigned int i = index; i < size; i += stride) {
            result[i] = (sharedArgMax[threadIdx.x] == idx) ? sharedDiffInput[threadIdx.x]
                                                           : 0.0f;
        }
    }
}