#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSSoftplus_propagate_kernel(float* x, float* y, unsigned int size)
{
    // Calculate global index
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop over the data using stride calculations for even data distribution amongst threads
    const unsigned int stride = blockDim.x * gridDim.x;
    for (unsigned int i = index; i < size; i += stride) {
        // Perform computation with conditional check to avoid unnecessary computation
        float exp_val = exp(x[i]);
        y[i] = exp_val > 0.0f ? log1pf(exp_val) : log1pf(0.0f + exp_val);
    }
}