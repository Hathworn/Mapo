#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSSqrt_kernel(unsigned int size, float* data)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory for better cache utilization
    extern __shared__ float shared_data[];
    
    for (unsigned int i = index; i < size; i += stride)
    {
        // Load data into shared memory
        shared_data[threadIdx.x] = data[i];
        __syncthreads();

        // Perform operation using shared memory
        shared_data[threadIdx.x] = sqrt(shared_data[threadIdx.x]);
        __syncthreads();

        // Write back the result to global memory
        data[i] = shared_data[threadIdx.x];
    }
}