#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaNoConversion_kernel(float * data, float * tickOutputsTraces, float * tickOutputsTracesLearning, float scaling, unsigned int inputDimX, unsigned int inputDimY, unsigned int inputDimZ)
{
    const unsigned int inputSize = inputDimX * inputDimY * inputDimZ;
    const unsigned int batchOffset = blockIdx.x * inputSize;
    const unsigned int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Process entire data set by utilizing all threads efficiently
    for (unsigned int idx = globalIdx; idx < inputSize; idx += gridDim.x * blockDim.x) {
        float value = data[idx + batchOffset];
        tickOutputsTraces[idx + batchOffset] = scaling * value;
        tickOutputsTracesLearning[idx + batchOffset] += scaling * value;
    }
}