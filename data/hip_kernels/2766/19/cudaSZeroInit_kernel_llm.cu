#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSZeroInit_kernel(unsigned int size, float* data)
{
    // Calculate a global index for each thread
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a single loop iteration without stride; better warp efficiency
    if (index < size)
    {
        data[index] = 0.0f;
    }
}