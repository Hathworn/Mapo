#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaUpdateFiringRate_kernel(int * firingRate, int * totalFiringRatePartial, unsigned int inputsDimX, unsigned int inputsDimY, unsigned int inputsDimZ)
{
    const unsigned int inputSize = inputsDimZ * inputsDimX * inputsDimY;
    const unsigned int batchInputOffset = blockIdx.z * inputSize;
    const unsigned int partialIdx = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ unsigned int partialSum[];

    // Initialize partial sum with 0, then add firingRate if within bounds
    partialSum[threadIdx.x] = (partialIdx < inputSize) ? firingRate[partialIdx + batchInputOffset] : 0;
    __syncthreads();

    // Optimized reduction using shared memory
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            partialSum[threadIdx.x] += partialSum[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Write the result of block-level reduction to global memory
    if (threadIdx.x == 0) {
        totalFiringRatePartial[blockIdx.x + gridDim.x * blockIdx.z] = partialSum[0];
    }
}