#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSRectifier_propagate_kernel(float* x, float* y, unsigned int size, float leakSlope, int shifting, float clipping)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use grid-stride loop for better parallelism
    for (unsigned int i = index; i < size; i += stride) {
        float value = x[i];

        // Use fast integer operations for shifting
        value = (shifting > 0) ? (value / (1 << shifting)) : ((shifting < 0) ? (value * (1 << (-shifting))) : value);

        // Combine conditional statements for efficiency
        y[i] = (value > 0.0f) ? ((clipping > 0.0f) ? min(value, clipping) : value) : (leakSlope * value);
    }
}