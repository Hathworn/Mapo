#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaUZeroInit_kernel(unsigned int size, unsigned int* data)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Unroll the loop to increase throughput and reduce loop overhead
    for (unsigned int i = index; i < size; i += stride * 4) {
        data[i] = 0U;
        if (i + stride < size) data[i + stride] = 0U;
        if (i + 2 * stride < size) data[i + 2 * stride] = 0U;
        if (i + 3 * stride < size) data[i + 3 * stride] = 0U;
    }
}