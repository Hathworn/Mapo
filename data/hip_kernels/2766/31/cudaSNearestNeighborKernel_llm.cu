#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSNearestNeighborKernel(const float* input, size_t inputSizeX, size_t inputSizeY, float* output, size_t outputSizeX, size_t outputSizeY, size_t nbChannels, size_t batchSize)
{
    const size_t inputOffset = (blockIdx.z * blockDim.z + threadIdx.z) * (nbChannels * inputSizeY * inputSizeX);
    const size_t outputOffset = (blockIdx.z * blockDim.z + threadIdx.z) * (nbChannels * outputSizeY * outputSizeX);

    const float multy = ((float) inputSizeY) / ((float) outputSizeY);
    const float multx = ((float) inputSizeX) / ((float) outputSizeX);

    size_t channel = blockIdx.x * blockDim.x + threadIdx.x; // Use threads for channels
    size_t oy = blockIdx.y * blockDim.y + threadIdx.y; // Use threads for output y-coordinates

    if (channel < nbChannels && oy < outputSizeY) {
        // Calculate input index
        size_t iy = static_cast<size_t>(oy * multy);
        float* outputPtr = output + outputOffset + channel * outputSizeY * outputSizeX + oy * outputSizeX;

        for(size_t ox = threadIdx.x; ox < outputSizeX; ox += blockDim.x * gridDim.x) { // Coalesce memory access for output
            size_t ix = static_cast<size_t>(ox * multx);

            outputPtr[ox] = input[inputOffset + channel * inputSizeY * inputSizeX + iy * inputSizeX + ix];
        }
    }
}