#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSadd_kernel(unsigned int size, float value, const float *x, float *y)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by checking bounds before accessing memory
    if (index < size) {
        y[index] = x[index] + value;
    }
}