#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSEuclideanSumBackward_kernel(unsigned int size, float* diffInput, float* input, float* output, const float scale, const float beta, float* result)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use a single loop with conditional operator for beta to avoid code duplication
    for (unsigned int i = index; i < size; i += stride) {
        if (output[i] != 0.0f) {
            result[i] = diffInput[i] * scale * (input[i] / output[i]) + beta * result[i];
        } else {
            result[i] *= beta;
        }
    }
}