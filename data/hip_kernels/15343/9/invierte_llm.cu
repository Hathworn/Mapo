#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void invierte(float *a, float *b) {
    // Calculate global thread ID using block and thread indices
    int id = threadIdx.x + blockDim.x * blockIdx.x;

    // Ensure global thread index is within bounds
    if (id < N) {
        // Reverse index access for array a
        b[id] = a[N - 1 - id];
    }
}