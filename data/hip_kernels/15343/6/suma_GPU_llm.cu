#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void suma_GPU(int a, int b, int *c)
{
    // Using shared memory for potential reuse
    __shared__ int temp;
    if (threadIdx.x == 0) {
        temp = a + b; // Perform computation
    }
    __syncthreads();
    
    // Assign result to output pointer
    if (threadIdx.x == 0) {
        *c = temp;
    }
}