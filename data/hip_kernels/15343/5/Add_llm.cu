#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Add(float *a, float *b, float *c)
{
    int Id = blockIdx.x * blockDim.x + threadIdx.x;
    if (Id < N) {
        a[Id] = threadIdx.x;  // Store thread index within block
        b[Id] = blockIdx.x;   // Store block index
        c[Id] = Id;           // Store global index
    }
}