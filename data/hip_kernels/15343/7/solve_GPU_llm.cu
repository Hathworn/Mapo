#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void solve_GPU(int a, int b, int c ,int *x1, int *x2)
{
    // Calculate common values once to avoid repeated computation
    int raiz = b * b - 4 * a * c;
    int i = -b;
    int j = 2 * a;
    float sqrt_raiz = sqrtf(raiz);

    // Compute the solutions using precomputed values
    *x1 = (i + sqrt_raiz) / j;
    *x2 = (i - sqrt_raiz) / j;
}