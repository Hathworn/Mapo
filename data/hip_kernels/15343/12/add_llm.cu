#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized with shared memory
__global__ void add(float *A, float *C)
{
    // Calculate column and row indices
    int columna = threadIdx.x;
    int fila = threadIdx.y;

    // Calculate linear index
    int Id = columna + fila * blockDim.x;

    // Define shared memory for data reuse
    extern __shared__ float tile[];

    // Load data into shared memory
    tile[threadIdx.y * blockDim.x + threadIdx.x] = A[Id];
    __syncthreads();

    // Calculate neighbor indices
    int id1 = (columna - 1) + fila * blockDim.x;
    int id2 = (columna + 1) + fila * blockDim.x;
    int id3 = columna + (fila - 1) * blockDim.x;
    int id4 = columna + (fila + 1) * blockDim.x;

    // Use shared memory when conditions are met
    if ((fila > 0 && fila < N - 1) && (columna > 0 && columna < N - 1)) {
        // Accumulate neighboring values using shared memory data
        C[Id] = tile[threadIdx.y * blockDim.x + (threadIdx.x - 1)] +
                tile[threadIdx.y * blockDim.x + (threadIdx.x + 1)] +
                tile[(threadIdx.y - 1) * blockDim.x + threadIdx.x] +
                tile[(threadIdx.y + 1) * blockDim.x + threadIdx.x];
    } else {
        // Boundary condition
        C[Id] = A[Id];
    }
}