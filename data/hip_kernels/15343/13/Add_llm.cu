#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Add(float *a, float *b, float *c, int n)
{
    int Id = threadIdx.x + blockDim.x * blockIdx.x;

    // Only perform computation if index is within bounds
    if (Id < n) {
        c[Id] = a[Id] * b[Id];
    }
}
```
