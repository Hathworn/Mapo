#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void suma(float *A, float *B, float *C)
{
    // Use blockIdx and blockDim to handle larger arrays efficiently
    int columna = threadIdx.x + blockIdx.x * blockDim.x;
    int fila = threadIdx.y + blockIdx.y * blockDim.y;
    int Id = columna + fila * (gridDim.x * blockDim.x);

    // Ensure index is within the bounds of the array
    if (columna < gridDim.x * blockDim.x && fila < gridDim.y * blockDim.y)
    {
        C[Id] = A[Id] + B[Id];
    }
}