#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CuKnlSetField(double xCells, double yCells, double* energy0, double* energy1)
{
    // Calculate unique thread index within grid
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate total number of threads in grid
    const int totalThreads = gridDim.x * blockDim.x;
    // Loop to ensure that every remaining element is processed
    for (int i = gid; i < xCells * yCells; i += totalThreads) {
        energy1[i] = energy0[i];
    }
}