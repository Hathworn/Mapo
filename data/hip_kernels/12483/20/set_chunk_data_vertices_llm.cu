#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize to use shared memory and avoid redundant calculations.
__global__ void set_chunk_data_vertices(int x, int y, int halo_depth, double dx, double dy, double x_min, double y_min, double* vertex_x, double* vertex_y, double* vertex_dx, double* vertex_dy) {
    extern __shared__ double shared_mem[];
    double* shared_vertex_x_min = shared_mem;
    double* shared_vertex_y_min = shared_mem + blockDim.x;
    
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(threadIdx.x == 0) {
        shared_vertex_x_min[0] = x_min - dx * halo_depth;
        shared_vertex_y_min[0] = y_min - dy * halo_depth;
    }
    
    __syncthreads();
    
    if (gid < x+1) {
        double local_x_min = shared_vertex_x_min[0];
        vertex_x[gid] = local_x_min + dx * gid;
        vertex_dx[gid] = dx; // Write once outside if condition
    }
    
    if (gid < y+1) {
        double local_y_min = shared_vertex_y_min[0];
        vertex_y[gid] = local_y_min + dy * gid;
        vertex_dy[gid] = dy; // Write once outside if condition
    }
}
```
