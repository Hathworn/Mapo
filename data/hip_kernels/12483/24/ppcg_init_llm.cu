#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ppcg_init(const int x_inner, const int y_inner, const int halo_depth, const double theta, const double* r, double* sd)
{
    // Calculate global thread index
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Early return if gid is out of bounds
    if (gid >= x_inner * y_inner) return;
    
    // Calculate x dimension including halos
    const int x = x_inner + 2 * halo_depth;
    
    // Use fast integer division by restricting access to grid dimensions
    const int col = gid % x_inner;
    const int row = gid / x_inner;
    
    // Compute offset using halo depth
    const int off0 = halo_depth * (x + 1);
    
    // Calculate actual array index leveraging pre-computed values
    const int index = off0 + col + row * x;
    
    // Perform division and store result in output array
    sd[index] = r[index] / theta;
}