#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void jacobi_init(const int x_inner, const int y_inner, const int halo_depth, const double* density, const double* energy, const double rx, const double ry, double* kx, double* ky, double* u0, double* u, const int coefficient)
{
    // Compute global thread ID
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= x_inner * y_inner) return;

    const int x = x_inner + 2 * halo_depth;
    const int col = gid % x_inner;
    const int row = gid / x_inner;
    const int off0 = halo_depth * (x + 1);
    const int index = off0 + col + row * x;

    // Precompute index-based accessed variables
    const double density_value = density[index];
    const double energy_value = energy[index];
    const double u_temp = energy_value * density_value;
    
    u0[index] = u_temp;
    u[index] = u_temp;

    // Early exit for halo region
    if (row == 0 || col == 0) return;

    // Variables to hold density values
    double density_center, density_left, density_down;

    // Precompute density values based on coefficient
    if (coefficient == CONDUCTIVITY) {
        density_center = density_value;
        density_left = density[index - 1];
        density_down = density[index - x];
    } else if (coefficient == RECIP_CONDUCTIVITY) {
        density_center = 1.0 / density_value;
        density_left = 1.0 / density[index - 1];
        density_down = 1.0 / density[index - x];
    }

    // Compute kx and ky
    const double density_left_center = density_left * density_center;
    const double density_down_center = density_down * density_center;
    kx[index] = rx * (density_left + density_center) / (2.0 * density_left_center);
    ky[index] = ry * (density_down + density_center) / (2.0 * density_down_center);
}