#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void finalise(const int x_inner, const int y_inner, const int halo_depth, const double* __restrict__ density, const double* __restrict__ u, double* __restrict__ energy)
{
    // Compute unique thread index for flattened 2D grid
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if thread is within bounds
    if (gid < x_inner * y_inner) {
        // Precompute constants outside loops for efficiency
        const int x = x_inner + 2 * halo_depth;
        const int off0 = halo_depth * (x + 1);
        
        // Determine 2D position within the block
        const int col = gid % x_inner;
        const int row = gid / x_inner;
        
        // Calculate global memory index
        const int index = off0 + col + row * x;

        // Finalize energy computation
        energy[index] = u[index] / density[index];
    }
}