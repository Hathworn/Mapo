#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unpack_bottom(const int x, const int y, const int halo_depth, double* field, double* buffer, const int depth)
{
    const int x_inner = x - 2 * halo_depth;

    // Use grid stride loop for better scalability
    for (int gid = threadIdx.x + blockDim.x * blockIdx.x; gid < x_inner * depth; gid += blockDim.x * gridDim.x) 
    {
        const int lines = gid / x_inner;
        const int offset = x * (halo_depth - depth) + lines * 2 * halo_depth;
        field[offset + gid] = buffer[gid];
    }
}