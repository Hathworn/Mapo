#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pack_bottom(const int x, const int y, const int halo_depth, double* field, double* buffer, const int depth)
{
    const int x_inner = x - 2 * halo_depth;

    // Calculate 2D index for improved memory coalescing
    int gid_x = threadIdx.x + blockIdx.x * blockDim.x;
    int gid_y = threadIdx.y + blockIdx.y * blockDim.y;

    // Check bounds in 2D space
    if (gid_x >= x_inner || gid_y >= depth) return;

    // Calculate global index in 1D
    int gid = gid_y * x_inner + gid_x; 

    // Compute field offset
    int offset = x * halo_depth + gid_y * 2 * halo_depth;
    buffer[gid] = field[offset + gid];
}