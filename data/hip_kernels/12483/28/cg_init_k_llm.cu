#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cg_init_k(const int x_inner, const int y_inner, const int halo_depth, const double* __restrict__ w, double* __restrict__ kx, double* __restrict__ ky, double rx, double ry)
{
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;
    if (gid >= x_inner * y_inner) return;

    const int x = x_inner + 2 * halo_depth - 1;
    const int col = gid % x_inner;
    const int row = gid / x_inner;
    const int off0 = halo_depth * (x + 1);
    const int index = off0 + col + row * x;

    double w_index = w[index]; // Cache repeated global memory access
    double w_index_1 = w[index - 1];
    double w_index_x = w[index - x];

    // Precompute reciprocal for reuse
    double rx_reciprocal = 1.0 / (2.0 * w_index * w_index_1);
    double ry_reciprocal = 1.0 / (2.0 * w_index * w_index_x);

    kx[index] = rx * (w_index_1 + w_index) * rx_reciprocal;
    ky[index] = ry * (w_index_x + w_index) * ry_reciprocal;
}