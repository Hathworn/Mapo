#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_u(const int x_inner, const int y_inner, const int halo_depth, const double* __restrict__ src, double* __restrict__ dest)
{
    // Use shared memory to optimize data access
    extern __shared__ double shared_src[];
    
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;
    const int x = x_inner + 2 * halo_depth;
    
    if (gid < x_inner * y_inner) {
        const int col = gid % x_inner;
        const int row = gid / x_inner;
        const int off0 = halo_depth * (x + 1);
        const int index = off0 + col + row * x;

        // Load data into shared memory
        shared_src[threadIdx.x] = src[index];
        __syncthreads();

        // Write data from shared memory to destination
        dest[index] = shared_src[threadIdx.x];
    }
}