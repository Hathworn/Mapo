#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ppcg_calc_sd(const int x_inner, const int y_inner, const int halo_depth, const double alpha, const double beta, const double* r, double* sd)
{
    // Calculate global thread ID
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure thread operates within data boundaries. Avoid out-of-bounds memory access
    if (gid >= x_inner * y_inner) return;

    // Calculate index considering the halo region
    const int x = x_inner + 2 * halo_depth;
    const int col = gid % x_inner;
    const int row = gid / x_inner;
    const int off0 = halo_depth * (x + 1);
    const int index = off0 + col + row * x;

    // Update sd at calculated index using alpha and beta
    sd[index] = alpha * sd[index] + beta * r[index];
}