#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_chunk_data(int x, int y, double dx, double dy, double* cell_x, double* cell_y, double* cell_dx, double* cell_dy, double* vertex_x, double* vertex_y, double* volume, double* x_area, double* y_area)
{
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use different variable to optimize logic separation and reduce checks
    int cellLimit = max(x, y);

    // Unroll conditions and use branching to reduce diverging
    if (gid < cellLimit) {
        if (gid < x) {
            cell_x[gid] = 0.5 * (vertex_x[gid] + vertex_x[gid + 1]);
            cell_dx[gid] = dx;
        }

        if (gid < y) {
            cell_y[gid] = 0.5 * (vertex_y[gid] + vertex_y[gid + 1]);
            cell_dy[gid] = dy;
        }
    }

    // Avoid redundant computation for each condition
    int xy = x * y;
    if (gid < xy) {
        volume[gid] = dx * dy;
    }

    int xyPlusY = (x + 1) * y;
    if (gid < xyPlusY) {
        x_area[gid] = dy;
    }

    int xTimesYPlusOne = x * (y + 1);
    if (gid < xTimesYPlusOne) {
        y_area[gid] = dx;
    }
}