#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function for unpack_top
__global__ void unpack_top(const int x, const int y, const int halo_depth, double* field, double* buffer, const int depth)
{
    // Calculate the index within the inner region
    const int x_inner = x - 2 * halo_depth;

    // Calculate the global thread ID
    const int gid = threadIdx.x + blockDim.x * blockIdx.x;

    // Return early if gid is out of bounds
    if (gid >= x_inner * depth) return;

    // Calculate line number and offset for field array
    const int lines = gid / x_inner;
    const int offset = x * (y - halo_depth) + lines * 2 * halo_depth;

    // Update the field with buffer value at calculated index
    field[offset + gid] = __ldg(&buffer[gid]); // Use __ldg to load buffer into cache
}