#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void jacobi_copy_u(const int x_inner, const int y_inner, const double* __restrict__ src, double* __restrict__ dest)
{
    const int gid = threadIdx.x + blockIdx.x * blockDim.x;

    // Unroll loop for better memory coalescing
    if (gid < x_inner * y_inner) {
        #pragma unroll
        for (int i = gid; i < x_inner * y_inner; i += blockDim.x * gridDim.x) {
            dest[i] = src[i];
        }
    }
}