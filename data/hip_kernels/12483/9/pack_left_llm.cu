#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pack_left(const int x, const int y, const int halo_depth, double* field, double* buffer, const int depth)
{
    const int y_inner = y - 2 * halo_depth;
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit for out-of-bounds threads
    if (gid >= y_inner * depth) return;

    // Calculate the line index
    const int line = gid / depth;
    
    // Compute the offset using line and thread index
    const int offset = halo_depth + line * x + (gid % depth);

    buffer[gid] = field[offset];
}