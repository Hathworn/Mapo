#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void unpack_left(const int x, const int y, const int halo_depth, double* __restrict__ field, const double* __restrict__ buffer, const int depth)
{
    const int y_inner = y - 2 * halo_depth;
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Terminate early if thread index exceeds bounds
    if (gid >= y_inner * depth) return;

    // Calculate index using divide operation once and offset with computed index directly
    const int offset_base = halo_depth - depth + (gid / depth) * (x - depth);
    field[offset_base + gid % depth] = buffer[gid];
}