#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pack_right(const int x, const int y, const int halo_depth, double* field, double* buffer, const int depth)
{
    // Efficient calculation of global thread ID
    const int gid = threadIdx.x + blockDim.x * blockIdx.x;
    const int y_inner = y - 2 * halo_depth;

    // Early exit for threads outside the bounds
    if (gid >= y_inner * depth) return;

    // Directly calculate the field index
    const int lines = gid / depth;
    const int offset = lines * x + halo_depth + depth - x;
    buffer[gid] = field[offset + gid];
}