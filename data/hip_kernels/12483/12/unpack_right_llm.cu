#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unpack_right(const int x, const int y, const int halo_depth, double* field, double* buffer, const int depth)
{
    const int y_inner = y - 2 * halo_depth;

    // Optimize index calculation for better performance and readability.
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < y_inner * depth) 
    {
        const int lines = gid / depth;
        const int offset = x - halo_depth + lines * (x - depth);
        field[offset + gid] = buffer[gid];
    }
}