#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void zero_buffer(const int x, const int y, double* buffer)
{
    // Calculate 2D grid and block indices to improve performance
    const int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate unique global index
    const int gid = iy * x + ix;

    // Ensure gid is within the bounds of the buffer
    if (ix < x && iy < y)
    {
        buffer[gid] = 0.0;
    }
}