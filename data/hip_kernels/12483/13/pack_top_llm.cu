#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pack_top(const int x, const int y, const int halo_depth, double* field, double* buffer, const int depth)
{
    // Calculate effective inner dimension length
    const int x_inner = x - 2 * halo_depth;

    // Calculate global thread index
    const int gid = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Ensure gid is within bounds
    if (gid >= x_inner * depth) return;

    // Calculate the number of complete lines processed
    const int lines = gid / x_inner;
    
    // Calculate offset with simplified index calculation
    const int offset = (x - halo_depth) + lines * (x - depth) + (gid % x_inner);
    buffer[gid] = field[offset];
}