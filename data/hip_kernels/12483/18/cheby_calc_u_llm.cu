#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cheby_calc_u(const int x_inner, const int y_inner, const int halo_depth, const double* p, double* u)
{
    // Calculate global thread index
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid >= x_inner * y_inner) return;

    // Calculate global memory index
    int x = x_inner + 2 * halo_depth;
    int col = gid % x_inner;
    int row = gid / x_inner;
    int index = halo_depth * (x + 1) + col + row * x;

    // Update u array
    u[index] += p[index];
}