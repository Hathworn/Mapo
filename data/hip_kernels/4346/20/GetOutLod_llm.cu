#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function for GetOutLod
__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod, const size_t lod_len, size_t* out_lod0) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < lod_len) {
        size_t lod_value = in_lod[index];  // Reduce memory accesses by using local variable
        out_lod0[index] = lod_value - num_erased[lod_value];
    }
}