#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void instance_iou_cuda_kernel(int64_t total_gt_instances, const int64_t* __restrict__ nInstance, int nProposal, const int64_t* __restrict__ proposals_idx, const int64_t* __restrict__ proposals_offset, const int64_t* __restrict__ instance_labels, const int64_t* __restrict__ offset_num_gt_instances, const int64_t* __restrict__ batch, const int64_t* __restrict__ instance_pointnum, float* proposals_iou)
{
    int proposal_id = blockIdx.x;
    if (proposal_id >= nProposal) return; // Eliminate unnecessary work.

    int start = proposals_offset[proposal_id];
    int end = proposals_offset[proposal_id + 1];
    int sampleIdx = batch[proposals_idx[start]];
    int sampleNInstances = nInstance[sampleIdx];
    int instanceOffset = offset_num_gt_instances[sampleIdx];
    int proposal_total = end - start;

    // Use shared memory to reduce global memory accesses for instance labels
    extern __shared__ int shared_mem[];
    int *shared_instance_labels = shared_mem;
  
    for (int i = threadIdx.x; i < end - start; i += blockDim.x) {
        shared_instance_labels[i] = instance_labels[proposals_idx[start + i]];
    }
    __syncthreads(); // Ensure all labels are loaded

    for (int instance_id = threadIdx.x; instance_id < sampleNInstances; instance_id += blockDim.x)
    {
        int instance_total = instance_pointnum[instanceOffset + instance_id];
        int intersection = 0;
        
        // Use shared memory to compare labels
        for (int i = 0; i < end - start; i++)
        {
            if (shared_instance_labels[i] == instance_id + 1) // 0 for "no instance"
            {
                intersection++;
            }
        }

        proposals_iou[instanceOffset + instance_id + proposal_id * total_gt_instances] =
            (float)intersection / ((float)(proposal_total + instance_total - intersection) + 1e-5);
    }
}
```
