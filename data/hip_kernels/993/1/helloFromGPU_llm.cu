#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU(void)
{
    // Optimizing to use the warp for efficient execution
    if (threadIdx.x == 5 && threadIdx.y == 0 && threadIdx.z == 0)
    {
        printf("Hello World from GPU thread %d!\n", threadIdx.x);
    }
}