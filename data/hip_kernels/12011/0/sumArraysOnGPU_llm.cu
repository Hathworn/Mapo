#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) { // Check bounds to avoid accessing out of array limits
        C[idx] = A[idx] + B[idx];
    }
}