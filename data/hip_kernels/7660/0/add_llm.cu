#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" 
__global__ void add(int n, float *a, float *b, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilizing grid-stride loop for scalability
    for(; i < n; i += blockDim.x * gridDim.x)
    {
        sum[i] = a[i] + b[i];
    }
}