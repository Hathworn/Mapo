#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void double_value(double *x, double *y)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    y[idx] = 2.0 * x[idx]; // Perform computation on global index
}