#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#include <hiprand_kernel.h>  // Ensure hiprand functions are available

__global__ void generate_initial_population(const unsigned int genes_len, unsigned char* all_genes, unsigned int seed) {
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global index
    hiprandState_t state;
    hiprand_init(seed, globalIndex, 0, &state);  // Initialize random state with offset 0

    int offset = globalIndex * genes_len;  // Pre-calculate memory offset
    for (int geneIndex = 0; geneIndex < genes_len; geneIndex++) {
        // Generate a random number and store directly into memory
        all_genes[offset + geneIndex] = hiprand(&state) % 256;
    }
}