#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"





__global__ void generate_initial_population(const unsigned int genes_len, unsigned char* all_genes, unsigned int seed) {
// cada thread se encarga de generarse a si mismo y colocarse en memoria principal de GPU.
int globalIndex = blockIdx.x * blockDim.x + threadIdx.x; // indice del individuo/thread
/* CUDA's random number library uses hiprandState_t to keep track of the seed value
we will store a random state for every thread  */
hiprandState_t state;
hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
(globalIndex + 1), /* the sequence number is only important with multiple cores */
genes_len, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
&state);

for (int geneIndex = 0; geneIndex < genes_len; geneIndex++) {
/* we have to initialize the state */

// se debe poder optimizar
/* hiprand works like rand - except that it takes a state as a parameter */
unsigned int result = hiprand(&state);
all_genes[geneIndex + globalIndex * genes_len] = (unsigned char) result
% 256; // write in global device memory
}
}