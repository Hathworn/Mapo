#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int fitness_fun(unsigned char* actual_genes, unsigned char* expected_genes, int genes_len) {
    if (actual_genes == NULL) {
        printf("actual genes are NULL\n");
        return 1;
    }
    if (expected_genes == NULL) {
        printf("expected_genes are NULL\n");
        return 2;
    }

    int match_counter = 0;
    for (int geneIndex = 0; geneIndex < genes_len; geneIndex++) {
        unsigned char a_byte = expected_genes[geneIndex];
        unsigned char b_byte = actual_genes[geneIndex];
        unsigned char res_byte = ~(a_byte ^ b_byte);

        unsigned int res_int = (unsigned int) res_byte;
        match_counter += __popc(res_int);
    }
    return match_counter;
}

__global__ void _fitness_test_kernel(unsigned char* expected_genes, unsigned char* actual_genes, const int genes_size, int* ftnss_ptr) {
    // Use single thread for computation
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        *ftnss_ptr = fitness_fun(actual_genes, expected_genes, genes_size);
    }
}