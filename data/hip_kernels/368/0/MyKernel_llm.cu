#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MyKernel(int *a, int *b, int *c, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure all threads within a warp reach this point before proceeding
    __syncthreads();

    if (idx < N) {
        // Use register for intermediate computation
        int temp = a[idx] + b[idx];
        c[idx] = temp;
    }
}