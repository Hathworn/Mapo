#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _GPU_Floyd_kernel(int k, int* G, int* P, int N) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col >= N) return;

    int row = blockIdx.y;
    int idx = N * row + col;

    // Use shared memory to store elements accessed frequently
    __shared__ int best;
    if (threadIdx.x == 0) {
        best = G[N * row + k];
    }
    __syncthreads();

    // Early exit if conditions not met
    if (best == INF || best > 10) return;

    int tmp_b = G[k * N + col];
    if (tmp_b == INF || tmp_b > 10) return;

    // Calculate the shortest path
    int cur = best + tmp_b;
    // Update if a shorter path is found
    if (cur < G[idx]) {
        G[idx] = cur;
        P[idx] = k;
    }
}