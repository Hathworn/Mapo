#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//function declaration
unsigned int getmax(unsigned int *, unsigned int);
//unsigned int getmaxSeq(unsigned int *, unsigned int);

__global__ void getmaxcu(unsigned int* num, int size, int threadCount)
{
    __shared__ int localBiggest[32];
    int tid = threadIdx.x; // Cache threadIdx.x
    int wid = tid / 32;    // Warp index for current thread

    if (tid < 32) {
        localBiggest[tid] = 0; // Initialize shared memory using first 32 threads
    }
    __syncthreads();

    int current = blockIdx.x * blockDim.x + tid; // Calculate global thread ID

    if (current < size) {
        atomicMax(&localBiggest[wid], num[current]); // Use atomicMax for correctness and performance within a warp
    }
    __syncthreads();

    if (tid == 0) {
        int biggest = localBiggest[0];
        for (int i = 1; i < 32; i++) {
            if (biggest < localBiggest[i]) {
                biggest = localBiggest[i]; // Find the largest number in this block
            }
        }
        num[blockIdx.x] = biggest; // Store result in global memory
    }
}