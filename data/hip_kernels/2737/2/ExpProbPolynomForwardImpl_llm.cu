#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ExpProbPolynomForwardImpl(const float* probs, int batchSize, const float* values, int polynomCount, int outputDim, float* out) {

    // Calculate indices for this thread
    int polynomId = blockIdx.x;
    const int dimId = blockIdx.y;
    int tid = threadIdx.x;

    // Ensure thread is within bounds
    if (tid >= batchSize) {
        return;
    }

    float sum = 0.0f;
    probs += tid; // Align probs pointer to current thread
    values += dimId; // Align values pointer to current dimension

    // Process polynomials
    while (polynomId < polynomCount) {
        // Use __ldg to cache reads from global memory
        const float polynomProb = __ldg(probs + polynomId * batchSize);
        const float valueOut = __ldg(values + polynomId * outputDim);
        // Accumulate results
        sum += polynomProb * valueOut;
        polynomId += gridDim.x; // Advance to the next polynomial
    }

    // Safely update the output using atomic addition
    atomicAdd(out + dimId * batchSize + tid, sum);
}