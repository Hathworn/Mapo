#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SigmoidProbPolynomProbsImpl(const float* features, int batchSize, const int* splits, const float* conditions, const int* polynomOffsets, int polynomCount, float lambda, float* probs) {
    // Use blockDim.x rather than batchSize to ensure full occupancy
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;
    
    while(threadId < batchSize) {
        const float* featurePtr = features + threadId;
        float* probPtr = probs + threadId;
        int polynomId = blockIdx.y;
        
        while (polynomId < polynomCount) {
            int offset = polynomOffsets[polynomId];
            int nextOffset = polynomOffsets[polynomId + 1];
            const int depth = nextOffset - offset;
            float logProb = 0;

            // Loop unrolling might aid in performance for small depth values
            for (int i = 0; i < depth; ++i) {
                const int f = __ldg(splits + offset + i);
                const float c = __ldg(conditions + offset + i);
                const float x = __ldg(featurePtr + f * batchSize);
                const float val = -lambda * (x - c);
                const float expVal = 1.0f + expf(val);
                const float isTrueLogProb = isfinite(expVal) ? log(expVal) : val;
                logProb -= isTrueLogProb;
            }

            const float prob = expf(logProb);
            probPtr[polynomId * batchSize] = prob;
            polynomId += gridDim.y;
        }
        threadId += step;
    }
}