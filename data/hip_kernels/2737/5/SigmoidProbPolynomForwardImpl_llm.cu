#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SigmoidProbPolynomForwardImpl(const float* probs, int batchSize, const float* values, int polynomCount, int outputDim, float* out) {
    
    int polynomId = blockIdx.x;
    const int dimId = blockIdx.y;
    int tid = threadIdx.x;

    if (tid >= batchSize) {
        return;
    }

    float sum = 0.0f;
    const float* probsLocal = probs + tid;
    const float* valuesLocal = values + dimId;

    // Unrolling loop for better performance
    while (polynomId < polynomCount) {
        float polynomProb = __ldg(probsLocal + polynomId * batchSize);
        float valueOut = __ldg(valuesLocal + polynomId * outputDim);
        sum += polynomProb * valueOut;
        polynomId += gridDim.x;
    }

    // Optimized atomic addition
    atomicAdd(out + dimId * batchSize + tid, sum);
}