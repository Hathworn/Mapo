#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ExpProbPolynomProbsImpl(const float* __restrict__ features, int batchSize, const int* __restrict__ splits, const float* __restrict__ conditions, const int* __restrict__ polynomOffsets, int polynomCount, float lambda, float* __restrict__ probs) {
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    int totalThreads = gridDim.x * blockDim.x;

    while (threadId < batchSize) {
        int polynomId = blockIdx.x;

        const float* feature_ptr = features + threadId;
        float* prob_ptr = probs + threadId;

        while (polynomId < polynomCount) {
            int offset = polynomOffsets[polynomId];
            int nextOffset = polynomOffsets[polynomId + 1];
            const int depth = nextOffset - offset;

            float logProb = 0.0f;
            bool zeroProb = false;

            for (int i = 0; i < depth && !zeroProb; ++i) {
                const int f = __ldg(splits + offset + i);
                const float c = __ldg(conditions + offset + i);
                const float x = __ldg(feature_ptr + f * batchSize);

                const float val = -lambda * x;
                const float expVal = 1.0f - expf(val);

                if (isfinite(log(expVal))) {
                    logProb += log(expVal);
                } else {
                    zeroProb = true;
                }
            }

            *prob_ptr = zeroProb ? 0.0f : expf(logProb);

            polynomId += gridDim.x;
            prob_ptr += batchSize;
        }
        threadId += totalThreads; // Ensure all threads can process different indices
    }
}