#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PolynomForwardImpl(const float* probs, int batchSize, const float* values, int polynomCount, int outputDim, float* out) {
    int polynomId = blockIdx.x;
    const int dimId = blockIdx.y;
    int tid = threadIdx.x;

    // Ensure each thread handles one element in the batch
    if (tid >= batchSize) {
        return;
    }

    float sum = 0;
    probs += tid;  // Align probs pointer with thread index
    values += dimId; // Align values pointer with dimension index
    
    // Optimize the loop using shared memory 
    __shared__ float sharedProbs[/* appropriate size here based on polynomId loop and block size */];
    __shared__ float sharedValues[/* appropriate size here based on polynomId loop and block size */];

    while (polynomId < polynomCount) {
        sharedProbs[tid] = __ldg(probs + polynomId * batchSize);
        sharedValues[tid] = __ldg(values + polynomId * outputDim);
        __syncthreads();  // Sync threads after loading data into shared memory
        
        sum += sharedProbs[tid] * sharedValues[tid]; // Use shared memory
        
        polynomId += gridDim.x;
        __syncthreads();  // Ensure all updates to shared memory are done before next iteration
    }
    
    atomicAdd(out + dimId * batchSize + tid, sum); // Atomic operation to avoid race conditions
}