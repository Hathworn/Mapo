#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory for better performance
    __shared__ float sharedAccuracy;

    // Load data into shared memory
    if (threadIdx.x == 0) {
        sharedAccuracy = *accuracy;
    }
    __syncthreads();

    // Perform division in shared memory
    if (threadIdx.x == 0) {
        sharedAccuracy /= N;
        *accuracy = sharedAccuracy; // Store result back to global memory
    }
}