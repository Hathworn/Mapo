#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread 0 to perform the division to avoid race conditions
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}