#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computePressure_CUDA(float* pressure, float* density, const int num, const float rho0, const float stiff)
{
    // Use blockIdx.x * blockDim.x + threadIdx.x for simpler index calculation
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within bounds
    if (i >= num) return;
    
    // Pre-compute reciprocal of rho0 for optimization
    float invRho0 = 1.0f / rho0;

    // Calculate pressure more efficiently
    pressure[i] = stiff * (powf(density[i] * invRho0, 7.0f) - 1.0f);

    // Clamp pressure to be non-negative
    if (pressure[i] < 0.0f) pressure[i] = 0.0f;
}