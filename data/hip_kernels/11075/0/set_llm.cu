#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define imin(a,b) (a<b?a:b)

__global__ void set(double *dx, int N)
{
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride to cover full data set
    int stride = blockDim.x * gridDim.x;
    
    for (int i = tid; i < N; i += stride)
    {
        dx[i] = 0.0; // Initialize array element
    }
}