#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prefix_sum_scan(uint* dev_main_array, uint* dev_auxiliary_array, const uint array_size) {
    // Compute unique global index
    const uint element = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within bounds and adjust index for second block onwards
    if (element < array_size) {
        const uint cluster_offset = dev_auxiliary_array[blockIdx.x]; // Use current block's offset
        dev_main_array[element + blockDim.x] += cluster_offset; // Adjust indexing by blockDim.x
    }
}