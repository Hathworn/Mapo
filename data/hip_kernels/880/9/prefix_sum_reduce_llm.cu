#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void down_sweep_512(uint* data_block) {
    for (uint i = 512; i >= 2; i >>= 1) {
        for (uint j = 0; j * blockDim.x + threadIdx.x < 512; ++j) {  // Simplified loop bounds
            const auto element = 511 - (j * blockDim.x + threadIdx.x) * i;
            if (element < 512) {
                const auto other_element = element - (i >> 1);
                const auto value = data_block[other_element];
                data_block[other_element] = data_block[element];
                data_block[element] += value;
            }
        }
        __syncthreads();
    }
}

__device__ void up_sweep_512(uint* data_block) {
    uint starting_elem = 1;
    for (uint i = 2; i <= 512; i <<= 1) {
        for (uint j = 0; j * blockDim.x + threadIdx.x < 512; ++j) {  // Simplified loop bounds
            const uint element = starting_elem + (j * blockDim.x + threadIdx.x) * i;
            if (element < 512) {
                data_block[element] += data_block[element - (i >> 1)];
            }
        }
        starting_elem += i;
        __syncthreads();
    }
}

__global__ void prefix_sum_reduce(uint* dev_main_array, uint* dev_auxiliary_array, const uint array_size) {
    __shared__ uint data_block[512];
    const uint last_block = array_size >> 9;

    if (blockIdx.x < last_block) {
        const uint first_elem = blockIdx.x << 9;
        data_block[threadIdx.x] = dev_main_array[first_elem + threadIdx.x];
        data_block[threadIdx.x + blockDim.x] = dev_main_array[first_elem + threadIdx.x + blockDim.x];
        __syncthreads();

        up_sweep_512(data_block);

        if (threadIdx.x == 0) {
            dev_auxiliary_array[blockIdx.x] = data_block[511];
            data_block[511] = 0;
        }
        __syncthreads();

        down_sweep_512(data_block);

        dev_main_array[first_elem + threadIdx.x] = data_block[threadIdx.x];
        dev_main_array[first_elem + threadIdx.x + blockDim.x] = data_block[threadIdx.x + blockDim.x];
        __syncthreads();
    } else {
        const auto elements_remaining = array_size & 0x1FF;
        if (elements_remaining > 0) {
            const auto first_elem = array_size - elements_remaining;
            data_block[threadIdx.x] = 0;
            data_block[threadIdx.x + blockDim.x] = 0;

            const auto elem_index = first_elem + threadIdx.x;
            if (elem_index < array_size) {
                data_block[threadIdx.x] = dev_main_array[elem_index];
            }
            if ((elem_index + blockDim.x) < array_size) {
                data_block[threadIdx.x + blockDim.x] = dev_main_array[elem_index + blockDim.x];
            }
            __syncthreads();

            up_sweep_512(data_block);

            if (threadIdx.x == 0) {
                dev_auxiliary_array[blockIdx.x] = data_block[511];
                data_block[511] = 0;
            }
            __syncthreads();

            down_sweep_512(data_block);

            if (elem_index < array_size) {
                dev_main_array[elem_index] = data_block[threadIdx.x];
            }
            if ((elem_index + blockDim.x) < array_size) {
                dev_main_array[elem_index + blockDim.x] = data_block[threadIdx.x + blockDim.x];
            }
        }
    }
}