#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory for faster access
    __shared__ float shared_accuracy;

    // Only one thread reads and divides accuracy
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy / N;
    }

    // Ensure all threads can safely read the shared value
    __syncthreads();

    // Assign the divided result back to the output
    if (threadIdx.x == 0) {
        *accuracy = shared_accuracy;
    }
}