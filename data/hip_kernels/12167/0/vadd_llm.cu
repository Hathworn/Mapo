#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{

} // extern "C"

__global__ void vadd(const float *a, const float *b, float *c)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check to prevent out-of-bounds memory access
    if (i < gridDim.x * blockDim.x) 
    {
        c[i] = a[i] + b[i];
    }
}