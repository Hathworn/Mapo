#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel with parallel reduction
__global__ void analyze(const float *input, float *sum, int numElements) {
    __shared__ float sharedSum[256]; // Allocate shared memory

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread computes its element if within bounds
    if (i < numElements) {
        sharedSum[tid] = input[i];
    } else {
        sharedSum[tid] = 0.0f; // Initialize out of bounds threads with zero
    }

    // Synchronize threads to ensure all writes to shared memory are done
    __syncthreads();

    // Perform parallel reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedSum[tid] += sharedSum[tid + stride];
        }
        __syncthreads();
    }

    // Write back result of reduction to global memory
    if (tid == 0) {
        atomicAdd(sum, sharedSum[0]); // Use atomic only for final sum
    }
}