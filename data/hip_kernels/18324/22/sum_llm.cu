#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum(const float *input, float *output, int numElements) {
    float val = 0.f;
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    // Process elements in parallel with stride
    for (int i = threadId; i < numElements; i += blockDim.x * gridDim.x) {
        val += input[i];
    }

    // Store partial result in shared memory
    __shared__ float sharedMemory[256];
    int laneId = threadIdx.x;
    sharedMemory[laneId] = val;

    // Ensure all warps have written their results
    __syncthreads();

    // Reduce within block
    if (laneId < 128) sharedMemory[laneId] += sharedMemory[laneId + 128];
    __syncthreads();
    if (laneId < 64) sharedMemory[laneId] += sharedMemory[laneId + 64];
    __syncthreads();
    if (laneId < 32) {
        sharedMemory[laneId] += sharedMemory[laneId + 32];
        sharedMemory[laneId] += sharedMemory[laneId + 16];
        sharedMemory[laneId] += sharedMemory[laneId + 8];
        sharedMemory[laneId] += sharedMemory[laneId + 4];
        sharedMemory[laneId] += sharedMemory[laneId + 2];
        sharedMemory[laneId] += sharedMemory[laneId + 1];
    }

    // Output result from thread 0 of each block
    if (laneId == 0) {
        atomicAdd(output, sharedMemory[0]);
    }
}