#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(uint32_t *v, uint32_t val, uint32_t n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        v[i] = val; // Assign value if within bounds
    }
    if (i == 0 && blockIdx.x == 0) {
        printf("init\n"); // Ensure printf is executed once
    }
}