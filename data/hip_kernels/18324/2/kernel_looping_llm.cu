#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_looping(float *point, unsigned int num) {
    // Calculate global index
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Calculate stride size for loops
    unsigned int stride = gridDim.x * blockDim.x;

    for (int iloop = 0; iloop < NLOOPS; ++iloop) {
        // Unroll loop to improve performance
        for (size_t offset = idx; offset < num; offset += stride) {
            point[offset] += 1;
        }
    }
}