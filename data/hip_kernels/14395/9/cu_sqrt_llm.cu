#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_sqrt(const float *A, float *B, const int n) {
    // Calculate global thread ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Use a loop to process multiple elements per thread
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        // Compute square root
        B[i] = sqrtf(A[i]);
    }
}
