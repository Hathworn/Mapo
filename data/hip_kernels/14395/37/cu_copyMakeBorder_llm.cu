#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_copyMakeBorder(const float *src, float* dst, const int rowssrc, const int colssrc, const int up, const int down, const int left, const int right, const int n) {
    // Assign a unique thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int colsdst = colssrc + left + right;

    // Loop over data with specified stride
    for(; tid < n; tid += stride) {
        int csrc = tid % colssrc;
        int rsrc = tid / colssrc;
        int rdst = up + rsrc;
        int cdst = left + csrc;

        // Copy source to destination with specified offset
        dst[rdst * colsdst + cdst] = src[tid];
    }
}