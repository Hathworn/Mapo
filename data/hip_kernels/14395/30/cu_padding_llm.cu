#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_padding(const float* src, float* dst, const int rows1, const int cols1, const int cols2, const int n) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Padding offset
    int pad = (cols2 - cols1) / 2;

    // Ensure thread works within bound
    if (tid < n) {
        int c1 = tid % cols1;
        int r1 = tid / cols1;
        int r2 = r1 + pad;
        int c2 = c1 + pad;

        // Write padded data
        dst[r2 * cols2 + c2] = src[tid];
    }
}