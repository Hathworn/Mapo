#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_drelu(const float* src, float* dst, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {  // Ensure thread within bounds
        float s = src[tid];
        dst[tid] = (s > 0.0f) ? 1.0f : 0.0f;  // Use ternary operator for conditional assignment
    }
}