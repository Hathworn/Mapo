#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_interpolation(const float* __restrict__ src, float* __restrict__ dst, const int colssrc, const int colsdst, const int _stride, const int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop unrolling for better performance
    for (int i = tid; i < n; i += stride) {
        int csrc = i % colssrc;
        int rsrc = i / colssrc;
        int rdst = rsrc * _stride;
        int cdst = csrc * _stride;
        dst[rdst * colsdst + cdst] = src[i];
    }
}