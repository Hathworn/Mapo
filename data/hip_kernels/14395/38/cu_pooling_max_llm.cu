#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_pooling_max(const float* __restrict__ src, float* __restrict__ dst, float* __restrict__ loc, const int rowssrc, const int colssrc, const int rowsdst, const int colsdst, const int stridex, const int stridey, const int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    while (tid < n) {
        int cdst = tid % colsdst;
        int rdst = tid / colsdst;
        int rsrc = rdst * stridey;
        int csrc = cdst * stridex;
        int xend = min(colssrc - 1, csrc + stridex - 1);
        int yend = min(rowssrc - 1, rsrc + stridey - 1);

        loc[tid] = (float)(rsrc * colssrc + csrc);
        float max_val = dst[tid];  // Cache the max value
		
        for (int i = rsrc; i <= yend; ++i) {
            for (int j = csrc; j <= xend; ++j) {
                float val = src[i * colssrc + j];  // Cache the current value
                if (val > max_val) {
                    max_val = val;
                    loc[tid] = (float)(i * colssrc + j);
                }
            }
        }
        dst[tid] = max_val;  // Store the max value back
        tid += stride;
    }
}