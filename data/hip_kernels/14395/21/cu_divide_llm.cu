#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_divide(const float* numerator, const float* denominator, float* dst, const int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < n; i += stride) {  // Use a for loop for better readability
        float denom_val = denominator[i];    // Load the denominator value once
        if (denom_val == 0.0f) {             // Use 0.0f for float comparisons
            dst[i] = 0.0f;
        } else {
            dst[i] = __fdividef(numerator[i], denom_val);
        }
    }
}