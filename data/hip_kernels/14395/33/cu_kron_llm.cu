#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_kron(const float *a, const float* b, float* dst, const int rowsa, const int colsa, const int rowsdst, const int colsdst, const int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int colsb = colsdst / colsa;
    int rowsb = rowsdst / rowsa;

    // Optimize by removing redundant calculations from inside the loop
    while (tid < n) {
        int c2 = tid % colsdst;
        int r2 = tid / colsdst;

        int rb = r2 % rowsb;     // Compute row index of matrix b
        int cb = c2 % colsb;     // Compute column index of matrix b
        int ra = r2 / rowsb;     // Compute row index of matrix a
        int ca = c2 / colsb;     // Compute column index of matrix a

        dst[tid] = a[ra * colsa + ca] * b[rb * colsb + cb];
        tid += stride;           // Move to next set of elements handled by this thread
    }
}