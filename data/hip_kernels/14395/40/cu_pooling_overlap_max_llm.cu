#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_pooling_overlap_max(const float* src, float* dst, float *loc, const int rowssrc, const int colssrc, const int rowsdst, const int colsdst, const int sizex, const int sizey, const int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    while (tid < n) {
        int cdst = tid % colsdst;
        int rdst = tid / colsdst;
        int rsrc = rdst;
        int csrc = cdst;
        float max_val = -FLT_MAX;  // Initialize to minimum float
        int max_loc = rsrc * colssrc + csrc;

        int xend = csrc + sizex - 1;
        int yend = rsrc + sizey - 1;

        for (int i = rsrc; i <= yend; ++i) {
            for (int j = csrc; j <= xend; ++j) {
                int idx = i * colssrc + j;
                if (src[idx] > max_val) {  // Check for max value
                    max_val = src[idx];
                    max_loc = idx;
                }
            }
        }

        dst[tid] = max_val;  // Set max value
        loc[tid] = (float)max_loc;  // Store location of max value
        tid += stride;
    }
}