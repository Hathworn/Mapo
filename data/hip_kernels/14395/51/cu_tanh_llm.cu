#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_tanh(const float* src, float* dst, const int n) {
    // Calculate the global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop with strided access to cover all elements
    for (int index = tid; index < n; index += blockDim.x * gridDim.x) {
        dst[index] = tanhf(src[index]);
    }
}