#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_relu(const float* src, float* dst, int n) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Unroll loop for efficiency
    for (int i = tid; i < n; i += stride) {
        // Use ternary operator for simplicity
        dst[i] = src[i] > 0.0f ? src[i] : 0.0f;
    }
}