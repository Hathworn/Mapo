#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_depadding(const float* src, float* dst, const int rows1, const int cols1, const int cols2, const int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int pad = (cols1 - cols2) / 2;

    // Use a fixed stride for better efficiency
    for (int idx = tid; idx < n; idx += blockDim.x * gridDim.x) {
        int c2 = idx % cols2;
        int r2 = idx / cols2;
        int r1 = r2 + pad;
        int c1 = c2 + pad;
        dst[idx] = src[r1 * cols1 + c1];
    }
}