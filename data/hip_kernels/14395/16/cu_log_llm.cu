#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_log(const float* __restrict__ src, float* __restrict__ dst, const int n) {
    // Optimize memory access by minimizing arithmetic operations
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    for (; tid < n; tid += stride) {
        dst[tid] = __logf(src[tid]);
    }
}