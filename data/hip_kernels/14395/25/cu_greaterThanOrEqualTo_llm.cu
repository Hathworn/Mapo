#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_greaterThanOrEqualTo(const float* __restrict__ src, float* __restrict__ dst, const float val, const int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Unroll loop for improved performance
    for (int i = tid; i < n; i += stride) {
        dst[i] = (src[i] >= val) ? 1.0f : 0.0f;  // Use ternary operator for conciseness
    }
}