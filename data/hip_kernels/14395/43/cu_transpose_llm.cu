#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_transpose(const float* src, float* dst, int colssrc, int colsdst, int n){
    // Calculate row and column for destination matrix
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        int cdst = tid % colsdst;
        int rdst = tid / colsdst;
        
        // Efficiently calculate indices for source matrix
        dst[tid] = src[cdst * colssrc + rdst];
    }
}