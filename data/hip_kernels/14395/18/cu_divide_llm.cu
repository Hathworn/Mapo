#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_divide(float *numerator, float denominator, const int n) {
    // Use a common pattern that calculates only within bounds to avoid unneeded checks
    for (int tid = threadIdx.x + blockIdx.x * blockDim.x; tid < n; tid += blockDim.x * gridDim.x) {
        numerator[tid] = __fdividef(numerator[tid], denominator);
    }
}