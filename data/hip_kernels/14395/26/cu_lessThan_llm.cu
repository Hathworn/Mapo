#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_lessThan(const float* __restrict__ src, float* __restrict__ dst, const float val, const int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    for(int i = tid; i < n; i += stride) {
        // Use branchless assignment for better performance
        dst[i] = (src[i] < val) ? 1.0f : 0.0f;
    }
}