#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_unpooling(const float* src, const float* loc, float* dst, const int colsdst, const int n) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize by ensuring that each thread only processes the necessary array index
    if(tid < n) {
        int loc_val = (int)loc[tid];
        int cdst = loc_val % colsdst;
        int rdst = loc_val / colsdst;
        dst[rdst * colsdst + cdst] = src[tid];
    }
}