#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_exp(const float* src, float* dst, const int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    // Unroll the loop for better performance
    for (int i = tid; i < n; i += stride) {
        dst[i] = __expf(src[i]);
    }
}