#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_sum(const float* src, float* sum, float *global_mem, const int n) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int local_tid = threadIdx.x;

    // load input into __shared__ memory
    float x = (tid < n) ? src[tid] : 0.0f;  // Use ternary to avoid branch divergence
    global_mem[local_tid] = x;
    __syncthreads();

    // contiguous range pattern
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (local_tid < offset) {
            global_mem[local_tid] += global_mem[local_tid + offset];
        }
        __syncthreads();  // Synchronize at the end of each reduction step
    }

    // thread 0 writes the final result
    if (local_tid == 0) {
        sum[blockIdx.x] = global_mem[0];
    }
}