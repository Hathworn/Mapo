#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_fliplr(const float* src, float* dst, const int rows, const int cols, const int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    // Calculate position one-time outside loop for efficiency
    int c = tid % cols;
    int r = tid / cols;
    
    while (tid < n) {
        // Directly compute source index instead of using a complex formula
        int src_index = (cols - c - 1) + r * cols;
        dst[tid] = src[src_index];
        tid += stride;
        
        // Update 'c' and 'r' based on new 'tid' value
        c = tid % cols;
        r = tid / cols;
    }
}