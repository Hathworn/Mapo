#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_equalTo(const float* __restrict__ src, float* __restrict__ dst, const float val, const int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use loop unrolling for better performance
    while (tid < n) {
        if (src[tid] == val) {
            dst[tid] = 1.0;
        } else {
            dst[tid] = 0.0;
        }
        tid += stride;

        if (tid < n) {
            if (src[tid] == val) {
                dst[tid] = 1.0;
            } else {
                dst[tid] = 0.0;
            }
            tid += stride;
        }

        if (tid < n) {
            if (src[tid] == val) {
                dst[tid] = 1.0;
            } else {
                dst[tid] = 0.0;
            }
            tid += stride;
        }

        if (tid < n) {
            if (src[tid] == val) {
                dst[tid] = 1.0;
            } else {
                dst[tid] = 0.0;
            }
            tid += stride;
        }
    }
}