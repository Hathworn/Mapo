#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 32

// Optimized global kernel function
__global__ void cu_multiply(const float* A, const float* B, float* C, int rowsa, int colsa, int rowsb, int colsb, int rowsc, int colsc) {
    __shared__ float sA[TILE_SIZE][TILE_SIZE];    // Tile size of 32x32
    __shared__ float sB[TILE_SIZE][TILE_SIZE];

    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    int Col = blockDim.x * blockIdx.x + threadIdx.x;
    float Cvalue = 0.0;

    for (int k = 0; k < (colsa + TILE_SIZE - 1) / TILE_SIZE; k++) {
        int tiledRow = threadIdx.y + k * TILE_SIZE;
        
        // Load A into shared memory
        if (Row < rowsa && tiledRow < colsa) {
            sA[threadIdx.y][threadIdx.x] = A[Row * colsa + tiledRow];
        } else {
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }

        // Load B into shared memory
        int tiledCol = threadIdx.y + k * TILE_SIZE;
        if (Col < colsb && tiledCol < rowsb) {
            sB[threadIdx.y][threadIdx.x] = B[tiledCol * colsb + Col];
        } else {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads();

        // Compute partial result
        for (int j = 0; j < TILE_SIZE; ++j) {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }

        __syncthreads();
    }

    // Store result
    if (Row < rowsc && Col < colsc) {
        C[Row * colsc + Col] = Cvalue;
    }
}