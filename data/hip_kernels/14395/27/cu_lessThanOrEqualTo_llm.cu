#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_lessThanOrEqualTo(const float* __restrict__ src, float* __restrict__ dst, const float val, const int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use loop unrolling for improved performance
    for (; tid < n; tid += stride * 4) {
        if (tid < n) {
            dst[tid] = (src[tid] <= val) ? 1.0f : 0.0f;
        }
        if (tid + stride < n) {
            dst[tid + stride] = (src[tid + stride] <= val) ? 1.0f : 0.0f;
        }
        if (tid + 2 * stride < n) {
            dst[tid + 2 * stride] = (src[tid + 2 * stride] <= val) ? 1.0f : 0.0f;
        }
        if (tid + 3 * stride < n) {
            dst[tid + 3 * stride] = (src[tid + 3 * stride] <= val) ? 1.0f : 0.0f;
        }
    }
}