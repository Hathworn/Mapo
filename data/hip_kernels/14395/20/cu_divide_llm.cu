#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_divide(const float numerator, const float* __restrict__ denominator, float* __restrict__ dst, const int n){
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    // Use loop unrolling to optimize iterations
    for (int i = tid; i < n; i += stride * 4) {
        if (i < n) {
            dst[i] = (denominator[i] != 0) ? __fdividef(numerator, denominator[i]) : 0.0f;
        }
        if (i + stride < n) {
            dst[i + stride] = (denominator[i + stride] != 0) ? __fdividef(numerator, denominator[i + stride]) : 0.0f;
        }
        if (i + 2 * stride < n) {
            dst[i + 2 * stride] = (denominator[i + 2 * stride] != 0) ? __fdividef(numerator, denominator[i + 2 * stride]) : 0.0f;
        }
        if (i + 3 * stride < n) {
            dst[i + 3 * stride] = (denominator[i + 3 * stride] != 0) ? __fdividef(numerator, denominator[i + 3 * stride]) : 0.0f;
        }
    }
}