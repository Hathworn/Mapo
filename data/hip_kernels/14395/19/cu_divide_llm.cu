#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_divide(const float* numerator, float* dst, const float denominator, const int n) {
    // Calculate thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use grid stride loop for efficient memory access
    for (int i = tid; i < n; i += blockDim.x * gridDim.x) {
        // Check denominator value and perform division
        dst[i] = (denominator == 0.0f) ? 0.0f : __fdividef(numerator[i], denominator);
    }
}