#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void cu_greaterThan(const float* __restrict__ src, float* __restrict__ dst, const float val, const int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Unroll loop for improved memory access optimization
    for (int i = tid; i < n; i += stride) {
        dst[i] = (src[i] > val) ? 1.0f : 0.0f;
    }
}