#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_pow(const float* src, float* dst, const float power, const int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    // Loop unrolling for improved performance
    for (int idx = tid; idx < n; idx += stride) {
        dst[idx] = powf(src[idx], power);
        if (idx + stride < n) dst[idx + stride] = powf(src[idx + stride], power);
        else break;
    }
}