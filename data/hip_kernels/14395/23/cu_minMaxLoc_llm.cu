#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_minMaxLoc(const float* src, float* minValue, float* maxValue, int* minLoc, int* maxLoc, float* minValCache, float* maxValCache, int*   minLocCache, int*   maxLocCache, const int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float val = (tid < n) ? src[tid] : src[0]; // Use ternary operator for boundary check
    int loc = (tid < n) ? tid : 0;            // Assign location only if within bounds
    
    maxValCache[threadIdx.x] = val;
    minValCache[threadIdx.x] = val;
    maxLocCache[threadIdx.x] = loc;
    minLocCache[threadIdx.x] = loc;
    __syncthreads();

    // Reducing within a block using shared memory
    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1){
        if(threadIdx.x < offset){
            // Max value and location reduction
            if(maxValCache[threadIdx.x] < maxValCache[threadIdx.x + offset]){
                maxValCache[threadIdx.x] = maxValCache[threadIdx.x + offset];
                maxLocCache[threadIdx.x] = maxLocCache[threadIdx.x + offset];
            }

            // Min value and location reduction
            if(minValCache[threadIdx.x] > minValCache[threadIdx.x + offset]){
                minValCache[threadIdx.x] = minValCache[threadIdx.x + offset];
                minLocCache[threadIdx.x] = minLocCache[threadIdx.x + offset];
            }
        }
        __syncthreads();
    }

    // Write result by the first thread in block
    if(threadIdx.x == 0){
        minValue[blockIdx.x] = minValCache[0];
        maxValue[blockIdx.x] = maxValCache[0];
        minLoc[blockIdx.x] = minLocCache[0];
        maxLoc[blockIdx.x] = maxLocCache[0];
    }
}