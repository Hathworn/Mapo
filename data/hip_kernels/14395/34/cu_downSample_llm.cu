#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_downSample(const float *src, float* dst, const int y_stride, const int x_stride, const int colssrc, const int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int colsdst = (colssrc + x_stride - 1) / x_stride;  // Simplified calculation of colsdst
    while (tid < n) {
        int cdst = tid % colsdst;
        int rdst = tid / colsdst;
        int rsrc = rdst * y_stride;
        int csrc = cdst * x_stride;
        dst[tid] = src[rsrc * colssrc + csrc];
        tid += stride;
    }
}