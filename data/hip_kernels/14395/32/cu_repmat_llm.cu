#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_repmat(const float *a, float* dst, const int rowsa, const int colsa, const int rowsdst, const int colsdst, const int n) {
    // Calculate the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Process multiple elements per thread
    for (int index = tid; index < n; index += blockDim.x * gridDim.x) {
        // Compute row and column mappings
        int c2 = index % colsdst;
        int r2 = index / colsdst;
        int ra = r2 % rowsa;
        int ca = c2 % colsa;
        // Perform the assignment
        dst[index] = a[ra * colsa + ca];
    }
}