#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_getRange(const float *src, float* dst, const int xstart, const int xend, const int ystart, const int yend, const int colssrc, const int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int colsdst = xend - xstart + 1;
    
    // Unroll the loop for improved memory coalescing
    for (; tid < n; tid += stride) {
        int cdst = tid % colsdst;
        int rdst = tid / colsdst;
        int rsrc = rdst + ystart;
        int csrc = cdst + xstart;
        dst[tid] = src[rsrc * colssrc + csrc];
    }
}