#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PadMatrixInLeadingDimensionKernel(const int8_t* __restrict__ src, int8_t* __restrict__ dst, int col_src, int col_dst) {
    // Calculate the global index for the thread
    int idx = blockIdx.x * col_src + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  // Total threads within the grid

    for (int i = idx; i < col_src * gridDim.x; i += stride) {
        // Copy element from source to destination with leading dimension padding
        dst[blockIdx.x * col_dst + (i % col_src)] = src[i];
    }
}