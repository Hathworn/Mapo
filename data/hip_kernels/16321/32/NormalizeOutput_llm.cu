#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Calculate the global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a conditional to avoid out-of-bound access
    if (idx < num_elements) {
        // Assign values to output array
        to_normalize[idx * 3] = batch_index;
        to_normalize[idx * 3 + 1] = class_index;
        to_normalize[idx * 3 + 2] = static_cast<int64_t>(original[idx]);
    }
}