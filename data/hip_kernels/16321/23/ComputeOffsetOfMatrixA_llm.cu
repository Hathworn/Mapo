#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global index
    if (idx < N) { // Ensure the global index is within bounds
        output[blockIdx.x * N + idx] = -col_sum[idx]; // Directly access using idx
    }
}