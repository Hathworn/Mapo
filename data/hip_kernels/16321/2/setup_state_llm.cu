#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Optimized to initialize the state per thread
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}