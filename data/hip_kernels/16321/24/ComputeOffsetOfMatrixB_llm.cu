#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) { // Ensure index is within bounds
        output[blockIdx.x * N + idx] = -row_sum[blockIdx.x]; // Linear indexing for better coalescing
    }
}