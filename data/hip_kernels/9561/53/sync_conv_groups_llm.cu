#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() {
    // Improved kernel with thread synchronization
    __syncthreads();  // Ensure all threads reach this point before proceeding
}