#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;
    
    // Loop unrolling for better memory coalescing and reducing loop overhead
    if (eidx < gridDim.x * DP_BLOCKSIZE) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE * 8) {
            sum += (eidx < numElements) ? a[eidx] * b[eidx] : 0.0f;
            if (eidx + gridDim.x * DP_BLOCKSIZE < numElements) sum += a[eidx + gridDim.x * DP_BLOCKSIZE] * b[eidx + gridDim.x * DP_BLOCKSIZE];
            if (eidx + 2 * gridDim.x * DP_BLOCKSIZE < numElements) sum += a[eidx + 2 * gridDim.x * DP_BLOCKSIZE] * b[eidx + 2 * gridDim.x * DP_BLOCKSIZE];
            if (eidx + 3 * gridDim.x * DP_BLOCKSIZE < numElements) sum += a[eidx + 3 * gridDim.x * DP_BLOCKSIZE] * b[eidx + 3 * gridDim.x * DP_BLOCKSIZE];
            if (eidx + 4 * gridDim.x * DP_BLOCKSIZE < numElements) sum += a[eidx + 4 * gridDim.x * DP_BLOCKSIZE] * b[eidx + 4 * gridDim.x * DP_BLOCKSIZE];
            if (eidx + 5 * gridDim.x * DP_BLOCKSIZE < numElements) sum += a[eidx + 5 * gridDim.x * DP_BLOCKSIZE] * b[eidx + 5 * gridDim.x * DP_BLOCKSIZE];
            if (eidx + 6 * gridDim.x * DP_BLOCKSIZE < numElements) sum += a[eidx + 6 * gridDim.x * DP_BLOCKSIZE] * b[eidx + 6 * gridDim.x * DP_BLOCKSIZE];
            if (eidx + 7 * gridDim.x * DP_BLOCKSIZE < numElements) sum += a[eidx + 7 * gridDim.x * DP_BLOCKSIZE] * b[eidx + 7 * gridDim.x * DP_BLOCKSIZE];
        }
    }
    
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Tree-based reduction
    if (threadIdx.x < 256) { shmem[threadIdx.x] += shmem[threadIdx.x + 256]; __syncthreads(); }
    if (threadIdx.x < 128) { shmem[threadIdx.x] += shmem[threadIdx.x + 128]; __syncthreads(); }
    if (threadIdx.x < 64)  { shmem[threadIdx.x] += shmem[threadIdx.x + 64];  __syncthreads(); }
    
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    // Write final result of this block's contribution
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}