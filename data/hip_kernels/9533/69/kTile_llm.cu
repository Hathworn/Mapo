#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate thread index and total number of threads
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    // Loop through target elements with stride equal to total number of threads
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Compute target y and x using division and modulus
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;

        // Calculate source indices using modulus for tiling
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Copy tiled values from source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}