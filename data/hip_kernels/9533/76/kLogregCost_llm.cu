#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = __float2int_rn(labels[tx]); // Fast conversion from float to int
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp); // Using fast intrinsic for log

        if (labelp != maxp) {
            correctProbs[tx] = 0.0f; // Ensure correct type consistency
        } else {
            // Use shared memory for reduction if necessary (based on occupancy needs)
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp);
            }
            correctProbs[tx] = 1.0f / __int2float_rn(numMax); // Fast conversion from int to float
        }
    }
}