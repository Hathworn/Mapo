#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    // Optimize by unrolling loops and handling more elements per thread
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = tid; i < N; i += stride) {
        ingrad[i] = indata[i] > 0 ? outgrad[i] : 0;
    }
}