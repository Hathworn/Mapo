#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateGradientsKernel(float4 *D, float4 *TD, unsigned int nVertices)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop manually for four iterations
    if (4*idx < nVertices) D[4*idx] = TD[4*idx];
    if (4*idx + 1 < nVertices) D[4*idx + 1] = TD[4*idx + 1];
    if (4*idx + 2 < nVertices) D[4*idx + 2] = TD[4*idx + 2];
    if (4*idx + 3 < nVertices) D[4*idx + 3] = TD[4*idx + 3];
}