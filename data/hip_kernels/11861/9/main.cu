#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "GradientAverageKernel.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float4 *D = NULL;
hipMalloc(&D, XSIZE*YSIZE);
float4 *TD = NULL;
hipMalloc(&TD, XSIZE*YSIZE);
unsigned int *NEIGHBOR = NULL;
hipMalloc(&NEIGHBOR, XSIZE*YSIZE);
unsigned int *NBOFFSETS = NULL;
hipMalloc(&NBOFFSETS, XSIZE*YSIZE);
unsigned int *nNeighbors = NULL;
hipMalloc(&nNeighbors, XSIZE*YSIZE);
unsigned int nVertices = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
GradientAverageKernel<<<gridBlock, threadBlock>>>(D,TD,NEIGHBOR,NBOFFSETS,nNeighbors,nVertices);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
GradientAverageKernel<<<gridBlock, threadBlock>>>(D,TD,NEIGHBOR,NBOFFSETS,nNeighbors,nVertices);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
GradientAverageKernel<<<gridBlock, threadBlock>>>(D,TD,NEIGHBOR,NBOFFSETS,nNeighbors,nVertices);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}