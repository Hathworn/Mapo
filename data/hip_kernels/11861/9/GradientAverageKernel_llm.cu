#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GradientAverageKernel(float4 *D, float4 *TD, unsigned int *NEIGHBOR, unsigned int *NBOFFSETS, unsigned int *nNeighbors, unsigned int nVertices)
{
    int offset, soffset;
    float4 nbd, td;

    // Optimized cache for 4 elements per thread
    __shared__ float4 SI[4 * BLOCK_SIZE_AVGG];

    int vidxb = 4 * (blockIdx.x * blockDim.x) + threadIdx.x;
    int bidx = 4 * threadIdx.x;

    // Efficiently load shared memory
    for (int vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_AVGG && vidx < nVertices; vidx += BLOCK_SIZE_AVGG)
    {
        SI[bidx] = D[vidx];
        bidx++;
    }

    __syncthreads();

    bidx = 4 * threadIdx.x;
    // Process vertices in the block
    for (int vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_AVGG && vidx < nVertices; vidx += BLOCK_SIZE_AVGG)
    {
        offset = NBOFFSETS[vidx];
        int N = nNeighbors[vidx];

        td = SI[bidx++];

        // Accumulate neighbor values
        for (int n = 0; n < N; n++)
        {
            soffset = NEIGHBOR[offset + n];
            nbd = D[soffset];

            td.x += nbd.x;
            td.y += nbd.y;
            td.z += nbd.z;
        }

        // Average the accumulated values
        td.x /= (float)(N + 1);
        td.y /= (float)(N + 1);
        td.z /= (float)(N + 1);

        TD[vidx] = td;
    }
}