#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeSphereVertexDistancesKernel(float4 *V, float *dist, unsigned int *NEIGHBOR, unsigned int *NBOFFSETS, unsigned int *nNeighbors, unsigned int nVertices, float circumference)
{
    int n, N;
    int offset, soffset;

    // Calculate the global vertex index
    int vidxb = 4 * (blockIdx.x * blockDim.x) + threadIdx.x;
    int basevert = 4 * (blockIdx.x * blockDim.x);

    // Create a cache for 4 elements per block (4*BLOCK_SIZE elements)
    __shared__ float4 SI[4 * BLOCK_SIZE_CVD];

    if (vidxb < nVertices) {
        // Load vertices into shared memory
        SI[threadIdx.x] = V[vidxb];
    }

    __syncthreads();

    if (vidxb < nVertices) {
        offset = NBOFFSETS[vidxb];
        N = nNeighbors[vidxb];
        float4 tv = SI[threadIdx.x];

        for (n = 0; n < N; n++)
        {
            soffset = NEIGHBOR[offset + n];
            float4 nv = (soffset >= basevert && soffset < basevert + 4 * BLOCK_SIZE_CVD) ? SI[soffset - basevert] : V[soffset];

            // Calculate dot product
            float dot = __fmaf_rn(tv.z, nv.z, __fmaf_rn(tv.y, nv.y, __fmul_rn(tv.x, nv.x)));

            // Calculate norms
            float n1 = __fmaf_rn(tv.z, tv.z, __fmaf_rn(tv.y, tv.y, __fmul_rn(tv.x, tv.x)));
            float n2 = __fmaf_rn(nv.z, nv.z, __fmaf_rn(nv.y, nv.y, __fmul_rn(nv.x, nv.x)));

            float norm = __fmul_rn(__frsqrt_rn(n1), __frsqrt_rn(n2));

            // Calculate distance
            if (norm < 1.0e-7f || fabsf(dot) > norm)
            {
                dist[offset + n] = 0.0f;
            }
            else
            {
                dist[offset + n] = __fmul_rn(circumference, acosf(dot * norm));
            }
        }
    }
}