#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    int batchIndex = blockIdx.x;
    int timeIndex = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;

    if (timeIndex < output_w) {
        // Directly calculate the offsets
        float *gradInput_data = gradInput + batchIndex * input_w * input_n + timeIndex * input_n * dW;
        float *gradOutput_data = gradOutput + batchIndex * output_w * input_n + timeIndex * input_n;
        float *indices_data = indices + batchIndex * output_w * input_n + timeIndex * input_n;

        // Use unrolled loop for memory coalescing optimization
        for (int feat = 0; feat < input_n; ++feat) {
            int gradInputIndex = static_cast<int>(indices_data[feat]) * input_n + feat;
            atomicAdd(&gradInput_data[gradInputIndex], gradOutput_data[feat]);
        }
    }
}