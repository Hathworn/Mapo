#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW) {
    // Compute output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;

    // Get weight
    float the_weight = weight[k];

    // Compute gradInput
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput * the_weight;

            // Pre-compute gradInput start position
            float *ptr_gradInputStart = gradInput + (yy * dH * input_w) + (xx * dW);

            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    // Use pre-computed gradInput start to optimize memory access
                    atomicAdd(&ptr_gradInputStart[kx], z);
                }
                ptr_gradInputStart += input_w;
            }
        }
    }
}