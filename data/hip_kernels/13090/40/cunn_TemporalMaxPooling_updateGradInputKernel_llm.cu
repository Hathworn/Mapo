#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Precalculate and streamline memory access
    int time_idx = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (time_idx < output_w) {
        int batch_offset = blockIdx.x * input_w * input_n;
        float *gradInput_data = gradInput + batch_offset + time_idx * input_n * dW;
        float *gradOutput_data = gradOutput + batch_offset + time_idx * input_n;
        float *indices_data = indices + batch_offset + time_idx * input_n;

        // Direct feature loop access
        for (int feat = 0; feat < input_n; ++feat) {
            int idx = (int)indices_data[feat] * input_n + feat;
            atomicAdd(&gradInput_data[idx], gradOutput_data[feat]); // Use atomic operation to avoid race conditions
        }
    }
}