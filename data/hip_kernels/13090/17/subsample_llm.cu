#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Deduce indices and strides
    int o = blockIdx.x;
    int k = o % input_n;
    int xx = threadIdx.x;
    int yy = blockIdx.y * blockDim.y + threadIdx.y;

    // Select input/output starting positions
    output += o * output_w * output_h;
    input += o * input_w * input_h;

    // Load weight and bias
    float the_weight = weight[k];
    float the_bias = bias[k];

    // Stride in blockDim.x allows the use of multiple threads in x direction
    for(; yy < output_h; yy += blockDim.y * gridDim.y) {
        for(int xx = threadIdx.x; xx < output_w; xx += blockDim.x) {
            float sum = 0;
            // Compute the mean of the input image...
            for(int ky = 0; ky < kH; ++ky) {
                for(int kx = 0; kx < kW; ++kx) {
                    sum += input[(yy * dH + ky) * input_w + (xx * dW + kx)];
                }
            }
            // Update output
            output[yy * output_w + xx] = the_weight * sum + the_bias;
        }
    }
}