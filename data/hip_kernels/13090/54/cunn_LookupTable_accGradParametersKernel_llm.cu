#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_LookupTable_accGradParametersKernel(float *input, float *indices, float *gradOutput, float *gradWeight, float *count, float defaultScale, long numel, long stride)
{
    int idx = blockIdx.x * 4 + threadIdx.y;  // Thread index within the grid
    const int SZ = 4;  // Number of operations each thread will perform

    // Check if the current input is the first of its group
    if (idx < numel && (idx == 0 || input[idx] != input[idx - 1]))
    {
        do
        {
            const int startFeature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
            const int weightRow = ((int)input[idx] - 1) * stride;
            const int gradOutputRow = ((int)indices[idx] - 1) * stride;
            const float scale = count ? defaultScale / count[idx] : defaultScale;

            float gradient[SZ];
            float weight[SZ];

            // Use a single loop to read gradOutput and gradWeight
            for (int ii = 0; ii < SZ; ii++)
            {
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride)
                {
                    gradient[ii] = gradOutput[gradOutputRow + featureDim];
                    weight[ii] = gradWeight[weightRow + featureDim];
                }
            }

            // Update weights using a single loop
            for (int ii = 0; ii < SZ; ii++)
            {
                weight[ii] += gradient[ii] * scale;
            }

            // Write updated weights back using a single loop
            for (int ii = 0; ii < SZ; ii++)
            {
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride)
                {
                    gradWeight[weightRow + featureDim] = weight[ii];
                }
            }

            idx++;
        } while (idx < numel && input[idx] == input[idx - 1]);
    }
}