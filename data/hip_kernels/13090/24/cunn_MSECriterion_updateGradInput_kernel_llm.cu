#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_MSECriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, float norm, int nframe, int dim)
{
    // Optimize using shared memory and unroll loop
    extern __shared__ float shared_input[];
    int k = blockIdx.x;
    float *gradInput_k = gradInput + k*dim;
    float *input_k = input + k*dim;
    float *target_k = target + k*dim;
    
    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;

    // Load inputs into shared memory
    for (int i=i_start; i<i_end; i+=i_step) {
        shared_input[threadIdx.x] = input_k[i];
        __syncthreads();
        
        // Perform computation using shared memory for better memory access pattern
        gradInput_k[i] = norm * (shared_input[threadIdx.x] - target_k[i]);
        __syncthreads();
    }
}