#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor)
{
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void upscale(float *input, float *output, long no_elements, int scale_factor, int d1, int d2, int d3)
{
    // Calculate global thread index more efficiently
    long ii = blockIdx.y * blockDim.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
    
    // Only process if within bounds
    if (ii >= no_elements) return;
    
    // Translate and copy value
    int ipidx = translate_idx(ii, d1, d2, d3, scale_factor);
    output[ii] = input[ipidx];
}