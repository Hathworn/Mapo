#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w) {
    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Pointers to the current output/input planes
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // Loop over the output dimensions
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        int y_start = static_cast<int>(floorf(static_cast<float>(yy) / output_h * input_h));

        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            int x_start = static_cast<int>(floorf(static_cast<float>(xx) / output_w * input_w));

            float *ptr_gradInput = gradInput + y_start * input_w + x_start;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float *ptr_ind_x = indices_x + yy * output_w + xx;
            float *ptr_ind_y = indices_y + yy * output_w + xx;

            // Load output gradient
            float z = *ptr_gradOutput; 

            // Calculate indices with bounds check
            int argmax_x = static_cast<int>(*ptr_ind_x) - 1;
            int argmax_y = static_cast<int>(*ptr_ind_y) - 1;

            // Accumulate the gradient value
            if (argmax_x >= 0 && argmax_x < input_w && argmax_y >= 0 && argmax_y < input_h) {
                ptr_gradInput[argmax_x + argmax_y * input_w] += z;
            }
        }
    }
}