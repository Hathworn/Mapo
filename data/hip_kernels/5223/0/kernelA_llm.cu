#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelA(float* r, float* x, float* y, float* z, int size)
{
    // Calculate the global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within bounds
    if (idx < size) {
      r[idx] = x[idx] * y[idx] + z[idx];
    }
}