#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure thread index is within array bounds
    if (i < size) {
        r[i] = x[i] * y[i] + z[i];
    }
}