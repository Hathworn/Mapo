#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bcnn_op_cuda_relu_grad_kernel(int n, float *x, float *dx)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation, remove unnecessary blockIdx.y and gridDim.x
    if (i < n) {
        dx[i] *= (x[i] > 0.0f); // Remove cast by using floating-point literal
    }
}