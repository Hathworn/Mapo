#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bcnn_op_cuda_relu_kernel(int n, float *x, float *y)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Improve efficiency by unrolling the loop
    for (; i < n; i += gridDim.x * blockDim.x) {
        y[i] = x[i] * (x[i] > 0);
    }
}