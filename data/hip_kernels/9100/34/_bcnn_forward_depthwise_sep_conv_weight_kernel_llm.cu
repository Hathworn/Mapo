#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_forward_depthwise_sep_conv_weight_kernel(int nthreads, float *src_data, float *weight_data, int channels, int dst_h, int dst_w, int src_h, int src_w, int kernel_sz, int stride, int pad, float *dst_data)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride_total = blockDim.x * gridDim.x;

    while (i < nthreads) {
        int n = i / channels / dst_h / dst_w;
        int c = (i / dst_h / dst_w) % channels;
        int h = (i / dst_w) % dst_h;
        int w = i % dst_w;
        
        float value = 0.0f;
        float *weight = weight_data + c * kernel_sz * kernel_sz;

        // Simplifying the loop by pre-computing input coordinates
        int h_start = -pad + h * stride;
        int w_start = -pad + w * stride;

        for (int kh = 0; kh < kernel_sz; ++kh) {
            int h_in = h_start + kh;
            if (h_in >= 0 && h_in < src_h) { // Check row validity
                for (int kw = 0; kw < kernel_sz; ++kw) {
                    int w_in = w_start + kw;
                    if (w_in >= 0 && w_in < src_w) { // Check column validity
                        int offset = ((n * channels + c) * src_h + h_in) * src_w + w_in;
                        value += weight[kw] * src_data[offset];
                    }
                }
            }
            weight += kernel_sz; // Move to the next row in weight data
        }
        dst_data[i] = value;
        i += stride_total; // Move to the next index managed by the thread
    }
}