#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bcnn_cuda_add_bias_kernel(float *output, float *bias, int num_channels, int spatial_size)
{
    // Calculate the unique index for each thread
    int idx = blockIdx.z * num_channels * spatial_size + blockIdx.y * spatial_size + threadIdx.x;
    
    // Ensure index is within spatial_size bounds
    if (threadIdx.x < spatial_size)
        output[idx] += bias[blockIdx.y];
}