#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory to optimize performance
__global__ void _bcnn_vsub_kernel(int n, float *a, float *b, float *y)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Ensure i is less than n
    if (i < n) {
        // Load data into shared memory
        extern __shared__ float shared_a[];
        extern __shared__ float shared_b[];

        shared_a[threadIdx.x] = a[i];
        shared_b[threadIdx.x] = b[i];

        // Synchronize to ensure complete loading
        __syncthreads();

        // Perform subtraction using shared memory
        y[i] = shared_a[threadIdx.x] - shared_b[threadIdx.x];
    }
}