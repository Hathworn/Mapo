#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bcnn_op_cuda_ramp_grad_kernel(int n, float *x, float *dx)
{
    // Calculate the global index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Improved bounds checking
    if (i < n) {
        // Use bool-to-float conversion
        dx[i] *= ((x[i] > 0.0f) + 0.1f);
    }
}