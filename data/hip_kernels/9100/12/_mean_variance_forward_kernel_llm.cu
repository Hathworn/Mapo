#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _mean_variance_forward_kernel(float *x, int b, int c, int wxh, float *mean, float *var)
{
    float scale = 1.0f / (b * wxh);
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit for threads with id >= c
    if (i >= c) return;

    // Initialize means and variances to zero
    float sum = 0.0f, sum_squares = 0.0f;
    
    // Compute mean and variance for each channel
    for (int j = 0; j < b; ++j) {
        for (int k = 0; k < wxh; ++k) {
            int ind = j * c * wxh + i * wxh + k;
            float val = x[ind];
            sum += val;
            sum_squares += val * val;
        }
    }

    // Normalize sum and sum of squares by scale
    mean[i] = sum * scale;
    var[i] = sum_squares * scale - mean[i] * mean[i];
}