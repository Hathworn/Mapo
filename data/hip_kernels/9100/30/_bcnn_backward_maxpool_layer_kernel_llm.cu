#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, float *diff, float *prev_delta, int *indexes)
{
    // Pre-compute constants
    int h = (in_h - 1) / stride + 1;
    int w = (in_w - 1) / stride + 1;
    int area = (size - 1) / stride;
    int w_offset = (-size - 1) / 2 + 1;
    int h_offset = (-size - 1) / 2 + 1;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) {
        return;
    }

    // Simplified index computation
    int j = id % in_w;
    int i = (id / in_w) % in_h;
    int k = (id / (in_w * in_h)) % in_c;
    int b = id / (in_w * in_h * in_c);

    float d = 0.0f;
    for (int l = -area; l <= area; ++l) {
        for (int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h) {
                int out_index = out_w + w * (out_h + h * (k + c * b));
                if (indexes[out_index] == id) {
                    d += diff[out_index];
                }
            }
        }
    }
    prev_delta[id] += d;
}