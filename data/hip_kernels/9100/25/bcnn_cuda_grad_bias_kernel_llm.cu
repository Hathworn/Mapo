#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bcnn_cuda_grad_bias_kernel(float *grad_bias, float *grad_data, int num_channels, int spatial_size)
{
    int channel = blockIdx.y; // Fetch channel once
    int batch_size = blockIdx.z; // Fetch batch size once
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Compute flat thread index

    if (idx < spatial_size) {
        atomicAdd(&grad_bias[channel], grad_data[(batch_size * num_channels + channel) * spatial_size + idx]); // Use atomicAdd for race condition handling
    }
}