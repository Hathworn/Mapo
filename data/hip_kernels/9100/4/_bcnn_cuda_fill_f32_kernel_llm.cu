#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_cuda_fill_f32_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use block and thread indexes to calculate the global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within bounds before performing computation
    if (idx < N) {
        X[idx * INCX] = ALPHA;
    }
}