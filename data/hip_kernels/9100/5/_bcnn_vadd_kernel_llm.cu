#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_vadd_kernel(int n, float *a, float *b, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Flattening computation of thread index
    while (i < n) { // Efficiently handle cases where n is greater than total threads
        y[i] = a[i] + b[i];
        i += blockDim.x * gridDim.x; // Use all available resources to cover large n
    }
}