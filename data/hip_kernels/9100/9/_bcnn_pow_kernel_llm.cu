#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_pow_kernel(int n, float *x, float a, float *y)
{
    // Use shared memory optimization and calculate global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        y[i] = powf(x[i], a); // Use powf for faster execution with single-precision floats
    }
}