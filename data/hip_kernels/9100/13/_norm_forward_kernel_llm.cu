#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _norm_forward_kernel(float *x, float *mean, float *variance, int b, int c, int wxh) {
    // Calculate index using a more efficient approach
    int ind = blockIdx.x * blockDim.x + threadIdx.x;

    if (ind >= b * c * wxh) return;

    // Precompute j index based on ind
    int j = (ind / wxh) % c;

    // Use a local variable for intermediate calculation
    float sqrt_var_epsilon = sqrt(variance[j] + 0.000001f);

    // Perform normalization
    x[ind] = (x[ind] - mean[j]) / sqrt_var_epsilon;
}