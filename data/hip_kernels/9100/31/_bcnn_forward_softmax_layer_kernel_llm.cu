#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_forward_softmax_layer_kernel(int n, int batch, float *input, float *output)
{
    float sum = 0.0f;
    float largest = -INFINITY;
    int b = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if this thread's task is in range
    if (b >= batch) return;

    // Calculate the offset for this batch
    int offset = b * n;

    // Find the largest value for numerical stability
    for (int i = 0; i < n; ++i) {
        float val = input[i + offset];
        largest = fmaxf(val, largest);
    }

    // Calculate the exponentials sum
    for (int i = 0; i < n; ++i) {
        sum += expf(input[i + offset] - largest);
    }

    // Calculate the normalization factor
    float norm_factor = (sum != 0.0f) ? (largest + logf(sum)) : (largest - 100.0f);

    // Normalize the softmax output
    for (int i = 0; i < n; ++i) {
        output[i + offset] = expf(input[i + offset] - norm_factor);
    }
}