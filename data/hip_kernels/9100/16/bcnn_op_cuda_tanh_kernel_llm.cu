#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bcnn_op_cuda_tanh_kernel(int n, float *x, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure calculations only performed within valid index range
    while (i < n) {
        y[i] = (exp(2 * x[i]) - 1) / (exp(2 * x[i]) + 1);
        i += gridDim.x * blockDim.x; // Utilize grid stride loop for parallelism improvement
    }
}