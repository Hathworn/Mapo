#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bcnn_op_cuda_clamp_grad_kernel(int n, float *x, float *dx)
{
    // Use blockDim.x and gridDim.x to calculate threadIndex for 1D block
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop over all elements with a stride to support large arrays
    for (; i < n; i += blockDim.x * gridDim.x) {
        // Optimize conditional operation with ternary operator
        dx[i] *= (x[i] > 0.0f && x[i] < 1.0f) ? 1.0f : 0.0f;
    }
}