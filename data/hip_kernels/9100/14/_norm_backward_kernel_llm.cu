#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _norm_backward_kernel(float *x, float *mean, float *var, float *mean_diff, float *var_diff, int b, int c, int wxh, float *grad)
{
    // Calculate index and check if within bounds
    int ind = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x * gridDim.x * blockIdx.y;
    if (ind >= b * c * wxh) return;

    // Precompute repetitive values outside the core computation
    int j = (ind / wxh) % c;
    float inv_sqrt_var = rsqrtf(var[j] + 0.00001f);
    float b_wxh_inv = 1.0f / (wxh * b);
    float x_mean_diff = x[ind] - mean[j];

    // Perform computation using precomputed values
    grad[ind] = grad[ind] * inv_sqrt_var + var_diff[j] * 2.0f * x_mean_diff * b_wxh_inv + mean_diff[j] * b_wxh_inv;
}