#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_add_scalar_kernel(int n, float a, float *y)
{
    // Use stride to cover all elements for better parallelism
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (; i < n; i += stride) {
        y[i] += a;
    }
}