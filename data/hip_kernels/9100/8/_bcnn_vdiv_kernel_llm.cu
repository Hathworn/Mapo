#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_vdiv_kernel(int n, float *a, float *b, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index using only blockIdx.x and threadIdx.x

    // Use a stride loop to allow grid-stride
    for (; i < n; i += blockDim.x * gridDim.x) {
        y[i] = a[i] / b[i];
    }
}