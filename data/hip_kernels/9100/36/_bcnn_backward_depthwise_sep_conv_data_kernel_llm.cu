#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_backward_depthwise_sep_conv_data_kernel(int nthreads, float *dst_grad, float *weight_data, int batch_size, const int channels, int dst_h, int dst_w, const int src_h, const int src_w, int kernel_sz, int stride, int pad, float *src_grad)
{
    // Compute thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop over work items assigned to this thread
    for (; i < nthreads; i += blockDim.x * gridDim.x) {
        int n = i / channels / src_h / src_w;
        int c = (i / src_h / src_w) % channels;
        int h = (i / src_w) % src_h;
        int w = i % src_w;
        float *weight = weight_data + c * kernel_sz * kernel_sz;
        float value = 0.0f;

        // Optimize inner loops by reducing constant operations
        for (int kh = 0; kh < kernel_sz; ++kh) {
            int h_out_s = h + pad - kh;
            if (h_out_s % stride == 0) {
                int h_out = h_out_s / stride;
                if (h_out >= 0 && h_out < dst_h) {
                    for (int kw = 0; kw < kernel_sz; ++kw) {
                        int w_out_s = w + pad - kw;
                        if (w_out_s % stride == 0) {
                            int w_out = w_out_s / stride;
                            if (w_out >= 0 && w_out < dst_w) {
                                // Calculate offset and accumulate result
                                int offset = ((n * channels + c) * dst_h + h_out) * dst_w + w_out;
                                value += weight[kw] * dst_grad[offset];
                            }
                        }
                    }
                }
            }
        }
        src_grad[i] += value;
    }
}