#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, float *input, float *output, int *indexes)
{
    int h = (in_h - 1) / stride + 1;
    int w = (in_w - 1) / stride + 1;
    int c = in_c;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) {
        return;
    }

    // Compute output feature map indices
    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    int out_index = j + w * (i + h * (k + c * b));
    float max = -INFINITY;
    int max_i = -1;

    // Unroll loops for better performance
    for (int l = 0; l < size; ++l) {
        for (int m = 0; m < size; ++m) {
            int cur_h = i * stride + l;
            int cur_w = j * stride + m;
            int valid = (cur_h < in_h) && (cur_w < in_w);

            if (valid) {
                int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
                float val = input[index];
                if (val > max) {
                    max = val;
                    max_i = index;
                }
            }
        }
    }
    output[out_index] = max;
    indexes[out_index] = max_i;
}