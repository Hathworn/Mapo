#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bcnn_op_cuda_ramp_kernel(int n, float *x, float *y)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Process elements within bounds only
    if (i < n) {
        y[i] = x[i] * (x[i] > 0) + 0.1f * x[i];
    }
}