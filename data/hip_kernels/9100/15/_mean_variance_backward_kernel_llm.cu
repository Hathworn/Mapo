#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _mean_variance_backward_kernel(float *x, float *grad, float *mean, float *var, int b, int c, int wxh, float *mean_diff, float *var_diff)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index is within valid range
    if (i >= c) return;

    // Use shared memory for efficient accumulation
    __shared__ float shared_mean_diff;
    __shared__ float shared_var_diff;
    if (threadIdx.x == 0) {
        shared_mean_diff = 0.0f;
        shared_var_diff = 0.0f;
    }
    __syncthreads();

    // Perform accumulation
    for (int j = 0; j < b; ++j) {
        for (int k = 0; k < wxh; ++k) {
            int ind = j * c * wxh + i * wxh + k;
            atomicAdd(&shared_mean_diff, grad[ind]); // Use atomicAdd for safe accumulation
            atomicAdd(&shared_var_diff, grad[ind] * (x[ind] - mean[i]));
        }
    }
    __syncthreads();

    // Compute final results
    if (threadIdx.x == 0) {
        mean_diff[i] = shared_mean_diff * (-1.0f / sqrt(var[i] + 0.00001f)); // Use thread 0 to write result
        var_diff[i] = shared_var_diff * -0.5f / (var[i] * sqrtf(var[i]) + 0.00001f);
    }
}