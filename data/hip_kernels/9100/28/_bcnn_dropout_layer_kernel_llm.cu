#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_dropout_layer_kernel(float *input, int size, float *rand, float prob, float scale)
{
    // Use blockIdx.z to get 3D grid support
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.y * blockDim.x + blockIdx.z * blockDim.x * gridDim.y * blockDim.y;
    if (id < size) {
        input[id] = (rand[id] < prob) ? 0 : input[id] * scale; // Apply dropout
    }
}
```
