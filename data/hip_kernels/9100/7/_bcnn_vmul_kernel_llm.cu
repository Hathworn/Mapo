#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void _bcnn_vmul_kernel(int n, float *a, float *b, float *y)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    int stride = blockDim.x * gridDim.x;           // Calculate stride for loop
    for (; i < n; i += stride) {
        y[i] = a[i] * b[i];                        // Process all elements in the range of n
    }
}