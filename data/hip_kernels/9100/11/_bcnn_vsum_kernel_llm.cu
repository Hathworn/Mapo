#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void _bcnn_vsum_kernel(int n, float *x, float *sum) {
    // Use shared memory for partial sums
    extern __shared__ float partial_sum[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    // Initialize shared memory
    partial_sum[tid] = (i < n) ? x[i] : 0.0f;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            partial_sum[tid] += partial_sum[tid + stride];
        }
        __syncthreads();
    }

    // Thread 0 writes the result to the output
    if (tid == 0) {
        atomicAdd(sum, partial_sum[0]);
    }
}