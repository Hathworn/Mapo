#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bcnn_op_cuda_tanh_grad_kernel(int n, float *x, float *dx)
{
    // Use parallel computation by each thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (; i < n; i += stride) {
        dx[i] *= (1 - x[i] * x[i]);
    }
}