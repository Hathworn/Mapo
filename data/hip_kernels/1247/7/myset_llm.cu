#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void myset(unsigned long long *p, unsigned long long v, long long n) {
    // Calculate global thread ID
    const long long tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread ID is within range and assign value
    if (tid < n) {
        p[tid] = v;
    }
}