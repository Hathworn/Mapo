#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_destinations(hiprandState *state, int n, const uint32_t *sources, uint32_t *destinations) {
    int first = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory to reduce access time for state
    __shared__ hiprandState shared_state;
    if (threadIdx.x == 0) {
        shared_state = state[first];
    }
    __syncthreads();
    hiprandState local_state = shared_state;

    // Loop unrolling for efficiency
    for (int id = first; id < n; id += stride * 4) {
        destinations[id] = sources[hiprand(&local_state) % n];
        if (id + stride < n) {
            destinations[id + stride] = sources[hiprand(&local_state) % n];
        }
        if (id + stride * 2 < n) {
            destinations[id + stride * 2] = sources[hiprand(&local_state) % n];
        }
        if (id + stride * 3 < n) {
            destinations[id + stride * 3] = sources[hiprand(&local_state) % n];
        }
    }

    // Restore updated state back to global memory
    if (threadIdx.x == 0) {
        state[first] = local_state;
    }
}