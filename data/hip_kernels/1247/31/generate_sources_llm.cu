#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_sources(hiprandState *state, int n, uint32_t *verts) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    if (id < n) {
        // Improved memory locality by keeping local state in register
        hiprandState local_state = state[id];
        
        // Using persistent threads to ensure no out-of-bounds access
        do {
            verts[id] = hiprand(&local_state);
            id += stride;
        } while (id < n);

        // Restore the state at the original position
        state[threadIdx.x + blockIdx.x * blockDim.x] = local_state;
    }
}