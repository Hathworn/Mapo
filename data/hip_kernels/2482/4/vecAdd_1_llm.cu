#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function for vector addition
__global__ void vecAdd_1(double *a, double *b, double *c, int n)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for loop

    for (int i = id; i < n; i += stride) // Loop over array elements with stride
    {
        c[i] = a[i] + b[i];
    }
}