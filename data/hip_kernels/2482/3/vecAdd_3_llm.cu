#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd_3(double *a, double *b, double *c, int n)
{
    // Calculate unique thread/global index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by iterating only if id is within bounds
    while (id < n * n)
    {
        c[id] = a[id] + b[id];
        
        // Increment id to process next element in this thread
        id += blockDim.x * gridDim.x;
    }
}