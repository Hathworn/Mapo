#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd_2(double *a, double *b, double *c, int n)
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop iteration optimized with stride access
    for (int i = id; i < n * blockDim.x; i += blockDim.x * gridDim.x) 
    {
        c[i] = a[i] + b[i];
    }
}