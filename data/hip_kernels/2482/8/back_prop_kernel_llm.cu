#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void back_prop_kernel(float *device_output, float *inP, float *m_hidden, float* weights_2, float* o_errG, int nInput, int nHidden, int nOutput, float l_R)
{
    int linearThreadIndex = threadIdx.x;
    int unit = blockIdx.x;

    // Use shared memory for weightedSum to improve access speed
    __shared__ float weightedSum;

    // Initialize shared memory
    if (linearThreadIndex == 0) {
        weightedSum = 0.0f;
    }
    __syncthreads();

    // Calculate weightedSum using all threads in the block
    for (int i = linearThreadIndex; i < nOutput; i += blockDim.x) {
        atomicAdd(&weightedSum, weights_2[unit * nOutput + i] * o_errG[i]);
    }
    __syncthreads();

    // Update only relevant device_output entries
    if (linearThreadIndex < nInput) {
        device_output[linearThreadIndex * nHidden + unit] = l_R * inP[linearThreadIndex] * m_hidden[unit] * (1 - m_hidden[unit]) * weightedSum;
    }
}