#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd_1(double *a, double *b, double *c, int n)
{
    // Use shared memory for reduced global memory access
    extern __shared__ double sharedMem[];

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Preload data into shared memory
    if (id < n) {
        sharedMem[tid] = a[id];
        __syncthreads(); // Ensure all threads load into shared memory

        // Perform addition using shared memory
        c[id] = sharedMem[tid] + b[id];
    }
}