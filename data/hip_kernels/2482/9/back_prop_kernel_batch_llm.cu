#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void back_prop_kernel_batch(float *device_output, float *inP, float *m_hidden, float* weights_2, float* o_errG, int nInput, int nHidden, int nOutput, float l_R, int batchSize)
{
    int linearThreadIndex = threadIdx.x;
    int unit = blockIdx.x % nHidden;
    int batch = blockIdx.x / nHidden;

    __shared__ float weightedSum;

    if (linearThreadIndex == 0 && unit < nHidden) {
        float sum = 0.0f;  // Initialize local sum
        for (int i = 0; i < nOutput; i++) {
            sum += weights_2[unit * nOutput + i] * o_errG[batch * (nOutput + 1) + i];
        }
        weightedSum = sum;  // Update shared memory with local sum
    }

    __syncthreads();

    if (linearThreadIndex < nInput) {
        float m_hidden_val = m_hidden[batch * (nHidden + 1) + unit];
        temp = l_R * inP[batch * (nInput + 1) + linearThreadIndex] * m_hidden_val * (1 - m_hidden_val) * weightedSum;

        atomicAdd(&device_output[linearThreadIndex * nHidden + unit], temp);
    }
}