#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd_3(double *a, double *b, double *c, int n)
{
    // Calculate unique global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by parallelizing outer loop across threads
    for(int i = id; i < n * n; i += blockDim.x * gridDim.x)
    {
        c[i] = a[i] + b[i];
    }
}