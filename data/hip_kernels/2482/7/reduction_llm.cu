#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(float *g_data, int n)
{
    __shared__ float s_data[NUM_ELEMENTS];

    int tid = threadIdx.x;
    int myIndex = threadIdx.x + blockIdx.x * blockDim.x;

    // Initialize shared memory with data from global memory
    s_data[tid] = (myIndex < n) ? g_data[myIndex] : 0.0f;

    __syncthreads();

    // Perform reduction using shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads();
    }

    // Write the result to global memory
    if (tid == 0)
    {
        g_data[blockIdx.x] = s_data[0];
    }
}