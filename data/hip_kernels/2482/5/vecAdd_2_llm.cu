#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd_2(double *a, double *b, double *c, int n)
{
    // Calculate global index for each thread
    int id_1 = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for larger arrays

    // Utilize stride to let each thread handle multiple elements
    for (int i = id_1; i < n * n; i += stride)
    {
        c[i] = a[i] + b[i];
    }
}