#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sobel(int width_d, int height_d, int threshold_d, unsigned int *pic_d, int *final_res) {
    int row_1 = blockIdx.y * blockDim.y + threadIdx.y;
    int col_1 = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.y;
    int ty = threadIdx.x;
    int width_Tile = TILE_SIZE;

    __shared__ int sharedTile[TILE_SIZE * TILE_SIZE];
    int magnitude, sum1, sum2;

    // Initialize shared memory tile
    if ((row_1 < height_d) && (col_1 < width_d)) {
        sharedTile[tx * width_Tile + ty] = pic_d[row_1 * width_d + col_1];
    } else {
        sharedTile[tx * width_Tile + ty] = 0;
    }
    __syncthreads();

    if (row_1 > 0 && col_1 > 0 && row_1 < height_d - 1 && col_1 < width_d - 1) {
        // Applying Sobel Filter using shared memory
        if ((tx > 0) && (tx < width_Tile - 1) && (ty > 0) && (ty < width_Tile - 1)) {
            sum1 = sharedTile[(tx-1) * width_Tile + (ty+1)] - sharedTile[(tx-1) * width_Tile + (ty-1)]
                 + 2 * sharedTile[tx * width_Tile + (ty+1)] - 2 * sharedTile[tx * width_Tile + (ty-1)]
                 + sharedTile[(tx+1) * width_Tile + (ty+1)] - sharedTile[(tx+1) * width_Tile + (ty-1)];

            sum2 = sharedTile[(tx-1) * width_Tile + (ty-1)] + 2 * sharedTile[(tx-1) * width_Tile + ty]
                 + sharedTile[(tx-1) * width_Tile + (ty+1)] - sharedTile[(tx+1) * width_Tile + (ty-1)]
                 - 2 * sharedTile[(tx+1) * width_Tile + ty] - sharedTile[(tx+1) * width_Tile + (ty+1)];

            magnitude = sum1 * sum1 + sum2 * sum2;

            final_res[row_1 * width_d + col_1] = (magnitude > threshold_d) ? 255 : 0;
        }
        __syncthreads();

        // Process pixels at the boundaries using global memory
        if ((tx == 0) || (ty == 0) || (tx == width_Tile - 1) || (ty == width_Tile - 1)) {
            if ((row_1 < height_d) && (col_1 < width_d)) {
                sum1 = pic_d[(row_1-1) * width_d + (col_1+1)] - pic_d[(row_1-1) * width_d + (col_1-1)]
                     + 2 * pic_d[row_1 * width_d + (col_1+1)] - 2 * pic_d[row_1 * width_d + (col_1-1)]
                     + pic_d[(row_1+1) * width_d + (col_1+1)] - pic_d[(row_1+1) * width_d + (col_1-1)];

                sum2 = pic_d[(row_1-1) * width_d + (col_1-1)] + 2 * pic_d[(row_1-1) * width_d + col_1]
                     + pic_d[(row_1-1) * width_d + (col_1+1)] - pic_d[(row_1+1) * width_d + (col_1-1)]
                     - 2 * pic_d[(row_1+1) * width_d + col_1] - pic_d[(row_1+1) * width_d + (col_1+1)];

                magnitude = sum1 * sum1 + sum2 * sum2;

                final_res[row_1 * width_d + col_1] = (magnitude > threshold_d) ? 255 : 0;
            }
        }
    }
}