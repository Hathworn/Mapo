#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void d_bucketsort(unsigned int * d_in, unsigned int * d_indices, unsigned int * d_sublist, unsigned int * r_outputlist, unsigned int * d_bucketoffsets, int itemCount) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < itemCount) {
        // Reduce redundant calculations of newpos within the loop
        unsigned int sublist_index = d_sublist[idx];
        unsigned int index_offset = d_indices[idx];
        int newpos = d_bucketoffsets[sublist_index] + index_offset;
        r_outputlist[newpos] = d_in[idx];
    }
}