#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sequence_gpu(int *d_ptr, int length)
{
    int elemID = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilizes cooperative threads to efficiently initialize indices
    if (elemID >= length) return;

    d_ptr[elemID] = elemID;
}