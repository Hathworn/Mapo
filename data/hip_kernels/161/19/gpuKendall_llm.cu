#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuKendall(const float * a, size_t na, const float * b, size_t nb, size_t sampleSize, double * results)
{
    size_t tx = threadIdx.x, ty = threadIdx.y,
           bx = blockIdx.x, by = blockIdx.y,
           rowa = bx * sampleSize, rowb = by * sampleSize;
    float concordant = 0.f;
    __shared__ float threadSums[NUMTHREADS * NUMTHREADS];

    for (size_t i = tx; i < sampleSize; i += NUMTHREADS) {
        for (size_t j = i + 1 + ty; j < sampleSize; j += NUMTHREADS) {
            concordant += (float)(((a[rowa + j] > a[rowa + i]) && (b[rowb + j] > b[rowb + i])) +
                                  ((a[rowa + j] < a[rowa + i]) && (b[rowb + j] < b[rowb + i])) +
                                  ((a[rowa + j] == a[rowa + i]) && (b[rowb + j] == b[rowb + i])));
        }
    }
    threadSums[ty * NUMTHREADS + tx] = concordant; // Store results in shared memory by transposed order
    __syncthreads();

    // Perform reduction within shared memory using transposed order for coalesced access
    for (size_t stride = NUMTHREADS / 2; stride > 0; stride >>= 1) {
        if (ty < stride) {
            threadSums[ty * NUMTHREADS + tx] += threadSums[(ty + stride) * NUMTHREADS + tx];
        }
        __syncthreads();
    }

    if (ty == 0) {
        // Further reduce across y-dimension
        atomicAdd(&threadSums[tx], threadSums[tx * NUMTHREADS]);
    }
    __syncthreads();

    if ((tx == 0) && (ty == 0)) {
        concordant = threadSums[0];
        float denom = (float)sampleSize;
        denom = (denom * (denom - 1.f)) / 2.f;
        float discordant = denom - concordant;
        float numer = concordant - discordant;
        results[by * na + bx] = ((double)numer) / ((double)denom);
    }
}