#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void noNAsPmccMeans(int nRows, int nCols, float * a, float * means)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if (col >= nCols) return; // Early exit if beyond column range

    int inOffset = col * nRows;
    float sum = 0.f;
    
    // Load data into shared memory for coalesced access
    __shared__ float threadSums[NUMTHREADS];
    
    for (int i = threadIdx.y; i < nRows; i += blockDim.y)
    {
        sum += a[inOffset + i];
    }

    threadSums[threadIdx.y] = sum; // Write per-thread sum to shared memory
    __syncthreads();

    // Reduce sum within block
    for (int stride = blockDim.y >> 1; stride > 0; stride >>= 1) {
        if (threadIdx.y < stride) {
            threadSums[threadIdx.y] += threadSums[threadIdx.y + stride];
        }
        __syncthreads();
    }
    
    // Write final result from block to global memory
    if (threadIdx.y == 0)
        means[col] = threadSums[0] / (float)nRows;
}