#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void maximum_kernel(const float *vg_a, size_t pitch_a, size_t n_a, const float *vg_b, size_t pitch_b, size_t n_b, size_t k, float *d, size_t pitch_d, float p) {
    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure the threads are within bounds
    if (x < n_a && y < n_b) {
        __shared__ float temp[NUM_THREADS * NUM_THREADS]; // Allocate sufficient shared memory
        float t_max = 0.0;

        // Avoid redundant threadIdx calculations
        temp[threadIdx.y * blockDim.x + threadIdx.x] = 0.0;

        for (size_t offset = threadIdx.y * blockDim.x + threadIdx.x; offset < k; offset += blockDim.x * blockDim.y) {
            float t = fabs(vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset]);
            t_max = max(t_max, t);
        }

        temp[threadIdx.y * blockDim.x + threadIdx.x] = t_max;

        // Sync within block
        __syncthreads();

        // Perform reduction in shared memory
        for (size_t stride = blockDim.x * blockDim.y / 2; stride > 0; stride >>= 1) {
            if (threadIdx.y * blockDim.x + threadIdx.x < stride) {
                temp[threadIdx.y * blockDim.x + threadIdx.x] = max(temp[threadIdx.y * blockDim.x + threadIdx.x], temp[threadIdx.y * blockDim.x + threadIdx.x + stride]);
            }
            __syncthreads();
        }

        // Store result in global memory
        if (threadIdx.y * blockDim.x + threadIdx.x == 0) {
            d[y * pitch_d + x] = temp[0];
        }
    }
}