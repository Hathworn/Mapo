#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512

// Updates the column norms by subtracting the Hadamard-square of the
// Householder vector.
//
// N.B.:  Overflow incurred in computing the square should already have
// been detected in the original norm construction.

__global__ void getColNorms(int rows, int cols, float *da, int lda, float *colNorms)
{
    int colIndex = threadIdx.x + blockIdx.x * blockDim.x;

    // Early exit if the column index exceeds the number of columns
    if (colIndex >= cols)
        return;

    float sum = 0.f;
    float *col = da + colIndex * lda;

    // Utilize shared memory to reduce global memory accesses
    __shared__ float sharedMem[512];

    for (int i = threadIdx.x; i < rows; i += blockDim.x) {
        float term = col[i];
        sharedMem[threadIdx.x] = term * term;
        __syncthreads();

        // Accumulate results from shared memory
        for (int j = 0; j < blockDim.x; j++) {
            sum += sharedMem[j];
        }
        __syncthreads();
    }

    colNorms[colIndex] = sum;
}