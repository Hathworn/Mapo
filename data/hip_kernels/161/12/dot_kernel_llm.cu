#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void dot_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p) {
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // If all element is to be computed
    if (x < n_a && y < n_b) {
        __shared__ float temp[NUM_THREADS];

        temp[threadIdx.x] = 0.0;
        // Loop unrolling for better performance
        size_t offset = threadIdx.x;
        while (offset + 3 < k) {
            temp[threadIdx.x] += vg_a[x * pitch_a + offset] * vg_b[y * pitch_b + offset];
            temp[threadIdx.x] += vg_a[x * pitch_a + offset + 1] * vg_b[y * pitch_b + offset + 1];
            temp[threadIdx.x] += vg_a[x * pitch_a + offset + 2] * vg_b[y * pitch_b + offset + 2];
            temp[threadIdx.x] += vg_a[x * pitch_a + offset + 3] * vg_b[y * pitch_b + offset + 3];
            offset += blockDim.x << 2;
        }
        for (; offset < k; offset++) {
            temp[threadIdx.x] += vg_a[x * pitch_a + offset] * vg_b[y * pitch_b + offset];
        }

        __syncthreads();

        // Reduce using shuffle for better performance
        float sum = temp[threadIdx.x];
        for (size_t stride = NUM_THREADS / 2; stride > 0; stride >>= 1) {
            sum += __shfl_down_sync(0xFFFFFFFF, sum, stride);
        }

        // Write to global memory
        if (threadIdx.x == 0) {
            d[y * pitch_d + x] = sum;
        }
    }
}