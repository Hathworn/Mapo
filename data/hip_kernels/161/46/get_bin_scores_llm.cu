#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float do_fraction(float numer, float denom) {
    float result = 0.f;

    if ((numer == denom) && (numer != 0.f))
        result = 1.f;
    else if (denom != 0.f)
        result = numer / denom;

    return result;
}

__global__ void get_bin_scores(int nbins, int order, int nknots, float* knots, int nsamples, int nx, float* x, int pitch_x, float* bins, int pitch_bins) {
    int col_x = blockDim.x * blockIdx.x + threadIdx.x;

    // Early exit if col_x is out of bounds
    if (col_x >= nx)
        return;

    float z, ld, rd, term1, term2;
    float* in_col = x + col_x * pitch_x;
    float* bin_col = bins + col_x * pitch_bins;
    int i0;

    for (int k = 0; k < nsamples; k++, bin_col += nbins) {
        z = in_col[k];
        i0 = (int)floorf(z) + order - 1;
        
        // Bounds correction
        i0 = min(max(i0, 0), nbins - 1);
        
        bin_col[i0] = 1.f;

        for (int i = 2; i <= order; i++) {
            for (int j = i0 - i + 1; j <= i0; j++) {
                if (j >= 0 && j < nbins && j < nknots && j + i - 1 >= 0) {
                    ld = do_fraction(z - knots[j], knots[j + i - 1] - knots[j]);
                    term1 = ld * bin_col[j];
                } else {
                    term1 = 0.f;
                }

                if (j + 1 >= 0 && j + 1 < nbins && j + 1 < nknots && j + i >= 0 && j + i < nknots) {
                    rd = do_fraction(knots[j + i] - z, knots[j + i] - knots[j + 1]);
                    term2 = rd * bin_col[j + 1];
                } else {
                    term2 = 0.f;
                }

                bin_col[j] = term1 + term2;
            }
        }
    }
}