#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuSDNoTest(const float *vectsA, size_t na, const float *vectsB, size_t nb, size_t dim, const float *means, const float *numPairs, float *sds)
{
    size_t
        offset, stride,
        tx = threadIdx.x,
        bx = blockIdx.x, by = blockIdx.y;
    float a, b, termA, termB;
    __shared__ float meanA, meanB, n;
    __shared__ float threadSumsA[NUMTHREADS], threadSumsB[NUMTHREADS];

    if ((bx >= na) || (by >= nb))
        return;

    if (tx == 0) {
        meanA = means[bx * nb * 2 + by * 2];
        meanB = means[bx * nb * 2 + by * 2 + 1];
        n = numPairs[bx * nb + by];
    }
    __syncthreads();

    threadSumsA[tx] = 0.f;
    threadSumsB[tx] = 0.f;
    
    for (offset = tx; offset < dim; offset += NUMTHREADS) {
        a = vectsA[bx * dim + offset];
        b = vectsB[by * dim + offset];

        termA = a - meanA;  // Calculate the difference from mean for vectsA
        termB = b - meanB;  // Calculate the difference from mean for vectsB
        threadSumsA[tx] += termA * termA;  // Sum squared differences for vectsA
        threadSumsB[tx] += termB * termB;  // Sum squared differences for vectsB
    }
    __syncthreads();

    // Reduce the sum of squares within the block using parallel reduction
    for (stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
        if (tx < stride) {
            threadSumsA[tx] += threadSumsA[tx + stride];  // Reduce sums for vectsA
            threadSumsB[tx] += threadSumsB[tx + stride];  // Reduce sums for vectsB
        }
        __syncthreads();
    }
    
    if (tx == 0) {
        sds[bx * nb * 2 + by * 2] = sqrtf(threadSumsA[0] / (n - 1.f));  // Calculate standard deviation for vectsA
        sds[bx * nb * 2 + by * 2 + 1] = sqrtf(threadSumsB[0] / (n - 1.f));  // Calculate standard deviation for vectsB
    }
}