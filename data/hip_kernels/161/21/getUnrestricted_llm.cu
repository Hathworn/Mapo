#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define max(a, b) ((a > b)?a:b)

#define THREADSPERDIM   16

#define FALSE 0
#define TRUE !FALSE

__global__ void getUnrestricted(int countx, int county, int rows, int cols, float *mX, int mXdim, float *vY, int vYdim, float *mQ, int mQdim, float *mR, int mRdim, float *vectB, int vectBdim) {
    int n = blockIdx.x * THREADSPERDIM + threadIdx.x;
    int m = blockIdx.y * THREADSPERDIM + threadIdx.y;
    if ((m >= county) || (n >= countx)) return;

    float* X = mX + (m * countx + n) * mXdim;
    float* Y = vY + (m * countx + n) * vYdim;
    float* Q = mQ + (m * countx + n) * mQdim;
    float* R = mR + (m * countx + n) * mRdim;
    float* B = vectB + (m * countx + n) * vectBdim;

    // Initialize the intercepts
    for (int i = 0; i < rows; i++) {
        X[i] = 1.f;
    }

    // Initialize Q with X ...
    for (int j = 0; j < cols; j++) {
        for (int i = 0; i < rows; i++) {
            Q[i + j * rows] = X[i + j * rows];
        }
    }

    // Gram-Schmidt process to find Q
    for (int j = 0; j < cols; j++) {
        float* colj = Q + rows * j;
        for (int i = 0; i < j; i++) {
            float* coli = Q + rows * i;
            float sum = 0.f;
            for (int k = 0; k < rows; k++) {
                sum += coli[k] * colj[k];
            }
            for (int k = 0; k < rows; k++) {
                colj[k] -= sum * coli[k];
            }
        }
        float sum = 0.f;
        for (int i = 0; i < rows; i++) {
            sum += colj[i] * colj[i];
        }
        float invnorm = 1.f / sqrtf(sum);
        for (int i = 0; i < rows; i++) {
            colj[i] *= invnorm;
        }
    }

    // Back substitution process
    for (int i = cols - 1; i > -1; i--) {
        float* colQ = Q + i * rows;
        // Matrix multiplication Q * X -> R
        for (int j = 0; j < cols; j++) {
            float* colX = X + j * rows;
            float sum = 0.f;
            for (int k = 0; k < rows; k++) {
                sum += colQ[k] * colX[k];
            }
            R[i + j * cols] = sum;
        }
        float sum = 0.f;
        // Compute the vector Q^t * Y -> B
        for (int j = 0; j < rows; j++) {
            sum += colQ[j] * Y[j];
        }
        // Back substitution to find the x for Rx = B
        for (int j = cols - 1; j > i; j--) {
            sum -= R[i + j * cols] * B[j];
        }
        B[i] = sum / R[i + i * cols];
    }
}