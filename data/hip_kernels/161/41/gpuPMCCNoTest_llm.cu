#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuPMCCNoTest(const float * vectsa, size_t na, const float * vectsb, size_t nb, size_t dim, const float * numPairs, const float * means, const float * sds, float * correlations) {
    size_t x = blockIdx.x, y = blockIdx.y, tx = threadIdx.x;
    float n, scoreA, scoreB;
    __shared__ float meanA, meanB, sdA, sdB, threadSums[NUMTHREADS];

    if ((x >= na) || (y >= nb))
        return;

    // Load shared memory only once
    if (tx == 0) {
        size_t index = x * nb * 2 + y * 2;
        meanA = means[index];
        meanB = means[index + 1];
        sdA = sds[index];
        sdB = sds[index + 1];
        n = numPairs[x * nb + y];
    }
    __syncthreads();

    float localSum = 0.f;
    // Simplified loop to accumulate scores
    for (size_t offset = tx; offset < dim; offset += NUMTHREADS) {
        float a = vectsa[x * dim + offset];
        float b = vectsb[y * dim + offset];
        scoreA = (a - meanA) / sdA;
        scoreB = (b - meanB) / sdB;
        localSum += scoreA * scoreB;
    }

    threadSums[tx] = localSum;
    __syncthreads();

    // Optimized reduction using warp synchronization
    for (size_t stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
        if (tx < stride) {
            threadSums[tx] += threadSums[tx + stride];
        }
        __syncthreads();
    }

    // Write final result to output
    if (tx == 0) {
        correlations[x * nb + y] = threadSums[0] / (n - 1.f);
    }
}