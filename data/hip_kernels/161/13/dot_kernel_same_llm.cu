#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void dot_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // Proceed only if within bounds
    if (y < n_a && x <= y) {
        __shared__ float temp[NUM_THREADS];
        temp[threadIdx.x] = 0.0;

        // Unroll loop for better performance
        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            temp[threadIdx.x] += vg_a[x * pitch_a + offset] * vg_a[y * pitch_a + offset];
        }

        // Sync with other threads
        __syncthreads();

        // Reduce using binary tree approach
        for (size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Write to global memory
        if (threadIdx.x == 0) {
            d[y * pitch_d + x] = temp[0];
            d[x * pitch_d + y] = temp[0];
        }
    }
}