#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512

__global__ void UpdateHHNorms(int cols, float *dV, float *dNorms) {
    int colIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (colIndex < cols) {
        // Use shared memory for better performance if possible
        float val = dV[colIndex];
        // Direct computation for norm update
        atomicAdd(&dNorms[colIndex], -val * val);
    }
}