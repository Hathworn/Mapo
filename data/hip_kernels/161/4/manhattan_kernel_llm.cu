#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void manhattan_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;
    
    // Check bounds once for all elements
    if(x < n_a && y < n_b) {
        __shared__ float temp[NUM_THREADS];
        
        temp[threadIdx.x] = 0.0;
        
        // Unroll the loop for better efficiency
        size_t offset = threadIdx.x;
        while(offset < k) {
            float t = fabs(vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset]);
            temp[threadIdx.x] += t;
            offset += blockDim.x;
        }

        // Sync with other threads
        __syncthreads();

        // Use reduction for summing
        for(size_t stride = blockDim.x / 2; stride > 0; stride /= 2) {
            if(threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Only first thread writes back result
        if(threadIdx.x == 0) {
            d[y * pitch_d + x] = temp[0];
        }
    }
}