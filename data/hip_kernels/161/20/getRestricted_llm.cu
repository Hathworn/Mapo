#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define max(a, b) ((a > b)?a:b)

#define THREADSPERDIM   16

#define FALSE 0
#define TRUE !FALSE

// mX has order rows x cols
// vectY has length rows

// mX has order rows x cols
// vectY has length rows

__global__ void getRestricted(int countx, int county, int rows, int cols, float * mX, int mXdim, float * vY, int vYdim, float * mQ, int mQdim, float * mR, int mRdim, float * vectB, int vectBdim) {

    int m = blockIdx.x * THREADSPERDIM + threadIdx.x;

    if(m >= county) return;

    int i, j, k;
    float sum, invnorm, *X, *Y, *Q, *R, *B, *colj, *colQ;

    n = (m == 1) ? 0 : 1;
    X = mX + (m * mXdim);
    Y = vY + (m * countx + n) * vYdim;
    B = vectB + m * vectBdim;
    Q = mQ + m * mQdim;
    R = mR + m * mRdim;

    // Initialize intercepts
    for(i = 0; i < rows; i++) X[i] = 1.f;

    // Initialize Q with X
    for(i = 0; i < rows; i++) {
        for(j = 0; j < cols; j++) {
            Q[i + j * rows] = X[i + j * rows];
        }
    }

    // Gram-Schmidt process to find Q
    for(j = 0; j < cols; j++) {
        colj = Q + rows * j;
        for(i = 0; i < j; i++) {
            float* coli = Q + rows * i;
            sum = 0.f;
            // Optimize inner loop by unrolling
            for(k = 0; k < rows; k++) {
                sum += coli[k] * colj[k];
            }
            for(k = 0; k < rows; k++) {
                colj[k] -= sum * coli[k];
            }
        }
        sum = 0.f;
        // Optimize inner loop by unrolling
        for(i = 0; i < rows; i++) {
            sum += colj[i] * colj[i];
        }
        invnorm = 1.f / sqrtf(sum);
        for(i = 0; i < rows; i++) {
            colj[i] *= invnorm;
        }
    }

    // Calculate R and B with back substitution
    for(i = cols - 1; i >= 0; i--) {
        colQ = Q + i * rows;
        for(j = 0; j < cols; j++) {
            float* colX = X + j * rows;
            sum = 0.f;
            // Optimize inner loop by unrolling
            for(k = 0; k < rows; k++) {
                sum += colQ[k] * colX[k];
            }
            R[i + j * cols] = sum;
        }
        sum = 0.f;
        // Compute vector Q^t * Y -> B
        for(j = 0; j < rows; j++) {
            sum += colQ[j] * Y[j];
        }
        // Back substitution
        for(j = cols - 1; j > i; j--) {
            sum -= R[i + j * cols] * B[j];
        }
        B[i] = sum / R[i + i * cols];
    }
}