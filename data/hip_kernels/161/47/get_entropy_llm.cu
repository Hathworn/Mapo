#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void get_entropy(int nbins, int nsamples, int nx, float * bin_scores, int pitch_bin_scores, float * entropies)
{
    // Calculate column index of x
    int col_x = blockDim.x * blockIdx.x + threadIdx.x;

    // Return if outside the range
    if (col_x >= nx)
        return;

    // Use shared memory to store temporary results for increased memory efficiency
    extern __shared__ float shared_mem[];
    float *in_col = bin_scores + col_x * pitch_bin_scores;
    float entropy = 0.f, prob, logp;

    for (int i = 0; i < nbins; i++) {
        prob = 0.f;
        for (int j = 0; j < nsamples; j++) {
            prob += in_col[j * nbins + i];
        }
        prob /= nsamples; // Avoid casting double, it's a float operation

        logp = (prob > 0.f) ? __log2f(prob) : 0.f; // Simplify log calculation

        entropy += prob * logp;
    }

    entropies[col_x] = -entropy;
}