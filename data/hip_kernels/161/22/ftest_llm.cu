#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define max(a, b) ((a > b)?a:b)

#define THREADSPERDIM   16

#define FALSE 0
#define TRUE !FALSE

__global__ void ftest(int diagFlag, int p, int rows, int colsx, int colsy, int rCols, int unrCols, float * obs, int obsDim, float * rCoeffs, int rCoeffsDim, float * unrCoeffs, int unrCoeffsDim, float * rdata, int rdataDim, float * unrdata, int unrdataDim, float * dfStats) 
{
    int j = blockIdx.x * THREADSPERDIM + threadIdx.x;
    int i = blockIdx.y * THREADSPERDIM + threadIdx.y;

    // Thread block out-of-bound check
    if (i >= colsy || j >= colsx) return;

    int idx = i * colsx + j;
    if (!diagFlag && i == j) {
        dfStats[idx] = 0.f;
        return;
    }

    float *tObs = obs + (i * colsx + j) * obsDim;
    float *tRCoeffs = rCoeffs + i * rCoeffsDim;
    float *tRdata = rdata + i * rdataDim;
    float *tUnrCoeffs = unrCoeffs + (i * colsx + j) * unrCoeffsDim;
    float *tUnrdata = unrdata + (i * colsx + j) * unrdataDim;

    float rSsq = 0.f, unrSsq = 0.f;
    float fp = (float)p, frows = (float)rows;

    // Unroll loop to improve performance
    for (int k = 0; k < rows; k += 2) {
        float unrEst1 = 0.f, rEst1 = 0.f, unrEst2 = 0.f, rEst2 = 0.f;
        float kobs1 = tObs[k], kobs2 = (k + 1 < rows) ? tObs[k + 1] : 0.f;

        for (int m = 0; m < rCols; m++) {
            rEst1 += tRCoeffs[m] * tRdata[k + m * rows];
            if (k + 1 < rows) rEst2 += tRCoeffs[m] * tRdata[k + 1 + m * rows];
        }
        for (int m = 0; m < unrCols; m++) {
            unrEst1 += tUnrCoeffs[m] * tUnrdata[k + m * rows];
            if (k + 1 < rows) unrEst2 += tUnrCoeffs[m] * tUnrdata[k + 1 + m * rows];
        }
        rSsq   += (kobs1 - rEst1) * (kobs1 - rEst1);
        unrSsq += (kobs1 - unrEst1) * (kobs1 - unrEst1);
        if (k + 1 < rows) {
            rSsq   += (kobs2 - rEst2) * (kobs2 - rEst2);
            unrSsq += (kobs2 - unrEst2) * (kobs2 - unrEst2);
        }
    }

    float score = ((rSsq - unrSsq) * (frows - 2.f * fp - 1.f)) / (fp * unrSsq);

    if (!isfinite(score))
        score = 0.f;

    dfStats[idx] = score;
}