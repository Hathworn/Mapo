#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuSignif(const float* gpuNumPairs, const float* gpuCorrelations, size_t n, float* gpuTScores)
{
    size_t bx = blockIdx.x;
    size_t tx = threadIdx.x;
    size_t start = bx * NUMTHREADS * THREADWORK + tx * THREADWORK;

    // Unroll loop for better performance
    #pragma unroll
    for (size_t i = 0; i < THREADWORK; i++) {
        size_t index = start + i;
        if (index < n) {
            float npairs = gpuNumPairs[index];
            float cor = gpuCorrelations[index];
            float radicand = (npairs - 2.f) / (1.f - cor * cor);
            gpuTScores[index] = cor * sqrtf(radicand);
        }
    }
}