#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void binary_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    if (x == y && x < n_a && threadIdx.x == 0) {
        d[y * pitch_d + x] = 0.0;
    }

    if (y < n_a && x < y) {
        __shared__ float temp[2 * NUM_THREADS];
        
        // Initialize shared memory once
        if (threadIdx.x < 2 * NUM_THREADS) {
            temp[threadIdx.x] = 0.0;
        }
        __syncthreads();
        
        // Calculate temp values across threads
        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            int a = vg_a[x * pitch_a + offset] != 0.0;
            int b = vg_a[y * pitch_a + offset] != 0.0;
            if (a ^ b) {
                atomicAdd(&temp[threadIdx.x], 1.0);
            }
            if (a || b) {
                atomicAdd(&temp[threadIdx.x + NUM_THREADS], 1.0);
            }
        }

        // Sync to ensure all threads have added their contributions
        __syncthreads();

        // Use first thread of the block to calculate the result
        if (threadIdx.x == 0) {
            float val = temp[0];
            if (temp[NUM_THREADS] != 0.0) {
                val /= temp[NUM_THREADS];
            }
            d[y * pitch_d + x] = val;
            d[x * pitch_d + y] = val;
        }
    }
}