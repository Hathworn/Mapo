#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuMeans(const float *vectsA, size_t na, const float *vectsB, size_t nb, size_t dim, float *means, float *numPairs)
{
    size_t bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x;
    float a, b;

    __shared__ float threadSumsA[NUMTHREADS], threadSumsB[NUMTHREADS], count[NUMTHREADS];

    if ((bx >= na) || (by >= nb))
        return;

    float sumA = 0.0f;  // Local variable for accumulated sum A
    float sumB = 0.0f;  // Local variable for accumulated sum B
    float cnt = 0.0f;  // Local variable for count

    for (size_t offset = tx; offset < dim; offset += NUMTHREADS) {
        a = vectsA[bx * dim + offset];
        b = vectsB[by * dim + offset];
        if (!(isnan(a) || isnan(b))) {
            sumA += a;
            sumB += b;
            cnt += 1.0f;
        }
    }

    // Store local sums into shared memory
    threadSumsA[tx] = sumA;
    threadSumsB[tx] = sumB;
    count[tx] = cnt;
    __syncthreads();

    // Reduce within block
    for (size_t stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
        if (tx < stride) {
            threadSumsA[tx] += threadSumsA[tx + stride];
            threadSumsB[tx] += threadSumsB[tx + stride];
            count[tx] += count[tx + stride];
        }
        __syncthreads();
    }

    // Write final result for block
    if (tx == 0) {
        means[bx * nb * 2 + by * 2] = threadSumsA[0] / count[0];
        means[bx * nb * 2 + by * 2 + 1] = threadSumsB[0] / count[0];
        numPairs[bx * nb + by] = count[0];
    }
}