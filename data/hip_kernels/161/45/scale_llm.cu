#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale(float knot_max, int nx, int nsamples, float* x, int pitch_x) {
    int col_idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Exit if column index is out of bounds
    if (col_idx >= nx) return;

    float min, max;
    float* col = x + col_idx * pitch_x;

    // Initialize min and max to the first element
    min = max = col[0];

    // Unroll the loop to find min and max
    for (int i = 1; i < nsamples; i++) {
        float val = col[i];
        if (val < min) min = val;
        if (val > max) max = val;
    }

    float delta = max - min;
    float inv_delta = knot_max / delta;  // Precompute inverse for efficiency

    // Scale the column
    for (int i = 0; i < nsamples; i++) {
        col[i] = (col[i] - min) * inv_delta;
    }
}