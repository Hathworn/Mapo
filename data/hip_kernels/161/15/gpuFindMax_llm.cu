#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512

__global__ void gpuFindMax(int n, float * data, int threadWorkLoad, int * maxIndex)
{
    int start = threadWorkLoad * threadIdx.x;
    int end = start + threadWorkLoad;
    __shared__ int maxIndicies[NTHREADS];

    maxIndicies[threadIdx.x] = -1;

    if(start >= n) return;

    int localMaxIndex = start;
    for(int i = start + 1; i < end && i < n; i++) { // Loop condition combined to improve readability
        if(data[i] > data[localMaxIndex])
            localMaxIndex = i;
    }
    maxIndicies[threadIdx.x] = localMaxIndex;
    __syncthreads();

    for(int i = blockDim.x / 2; i > 0; i /= 2) { // Use division for better clarity
        if(threadIdx.x < i) {
            int j = maxIndicies[threadIdx.x];
            int k = maxIndicies[i + threadIdx.x];
            if((k != -1) && (data[j] < data[k])) // Simplified condition check
                maxIndicies[threadIdx.x] = k;
        }
        __syncthreads();
    }
    if(threadIdx.x == 0) {
        *maxIndex = maxIndicies[0];
        // debug printing
        // printf("max index: %d\n", *maxIndex);
        // printf("max norm: %f\n", data[*maxIndex]);
        // end debug printing
    }
}