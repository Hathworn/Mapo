#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512

__global__ void makeHVector(int rows, float *input, float *output) {
    float elt, sum = 0.f; 
    extern __shared__ float sharedMem[];  // Use dynamic shared memory
    float* sums = sharedMem;
    __shared__ float beta;

    int tid = threadIdx.x, lane = threadIdx.x % warpSize; 
    for (int i = tid; i < rows; i += blockDim.x) {
        elt = input[i];
        output[i] = elt;
        sum += elt * elt;
    }
    sums[tid] = sum;
    __syncthreads();

    // In-warp reduction
    for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
        sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
    }

    if (lane == 0) {
        sums[tid / warpSize] = sum;
    }
    __syncthreads();

    // Cross-warp reduction
    if (tid < (blockDim.x / warpSize)) {
        sum = (tid < (rows + warpSize - 1) / warpSize) ? sums[tid] : 0;
        for (int offset = (blockDim.x / warpSize) / 2; offset > 0; offset >>= 1) {
            sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
        }
    }

    if (tid == 0) {
        elt = input[0];
        float norm = sqrtf(elt * elt + sum);
        elt = (elt > 0) ? elt + norm : elt - norm;
        output[0] = elt;
        beta = sqrtf(2.f / (elt * elt + sum));
    }
    __syncthreads();

    for (int i = tid; i < rows; i += blockDim.x) {
        output[i] *= beta;
    }
}