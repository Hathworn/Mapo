#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void canberra_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // Initialize diagonal elements to zero
    if (x == y && x < n_a && threadIdx.x == 0) {
        d[y * pitch_d + x] = 0.0;
    }

    // Compute for all elements where y < n_a and x < y
    if (y < n_a && x < y) {
        __shared__ float temp[NUM_THREADS];

        // Initialize shared memory
        temp[threadIdx.x] = 0.0;

        // Loop unrolling for better performance
        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float num = abs(vg_a[x * pitch_a + offset] - vg_a[y * pitch_a + offset]);
            float den = abs(vg_a[x * pitch_a + offset] + vg_a[y * pitch_a + offset]);
            if (den != 0.0) {
                temp[threadIdx.x] += num / den;
            }
        }

        // Synchronize threads
        __syncthreads();

        // Reduce with optimization
        for (size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Store result in global memory
        if (threadIdx.x == 0) {
            float s = temp[0];
            // Write symmetric elements to avoid recomputation
            d[y * pitch_d + x] = s;
            d[x * pitch_d + y] = s;
        }
    }
}