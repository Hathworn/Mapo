#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512

__global__ void gpuSwapCol(int rows, float * dArray, int coli, int * dColj, int * dPivot)
{
    int rowIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(rowIndex < rows) { // Simplified boundary check
        int colj = coli + (*dColj); // Compute target column index
        float fholder = dArray[rowIndex+coli*rows]; // Use temporary for swap
        dArray[rowIndex+coli*rows] = dArray[rowIndex+colj*rows];
        dArray[rowIndex+colj*rows] = fholder;
    }

    // Handle pivot swapping using thread 0 in block 0
    if(rowIndex == 0) {
        int iholder = dPivot[coli];
        dPivot[coli] = dPivot[colj];
        dPivot[colj] = iholder;
    }
}