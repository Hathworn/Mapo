#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

// Optimized kernel function
__global__ void gpuMeansNoTest(const float *vectsA, size_t na, const float *vectsB, size_t nb, size_t dim, float *means, float *numPairs) {
    size_t bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x;
    float a, b;

    __shared__ float threadSumsA[NUMTHREADS], threadSumsB[NUMTHREADS], count[NUMTHREADS];

    // Check if the block indices are in range
    if ((bx >= na) || (by >= nb)) return;

    // Initialize shared memory
    threadSumsA[tx] = 0.f;
    threadSumsB[tx] = 0.f;
    count[tx] = 0.f;

    // Improve memory coalescing by iterating with stride
    for (size_t offset = tx; offset < dim; offset += NUMTHREADS) {
        a = vectsA[bx * dim + offset];
        b = vectsB[by * dim + offset];

        threadSumsA[tx] += a;
        threadSumsB[tx] += b;
        count[tx] += 1.f;
    }
    __syncthreads();

    // Reduce with a single loop, combining into fewer instructions
    for (size_t stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
        if (tx < stride) {
            threadSumsA[tx] += threadSumsA[tx + stride];
            threadSumsB[tx] += threadSumsB[tx + stride];
            count[tx] += count[tx + stride];
        }
        __syncthreads();
    }

    // Atomic operations for writing results to global memory by the first thread
    if (tx == 0) {
        means[bx * nb * 2 + by * 2] = threadSumsA[0] / count[0];
        means[bx * nb * 2 + by * 2 + 1] = threadSumsB[0] / count[0];
        numPairs[bx * nb + by] = count[0];
    }
}