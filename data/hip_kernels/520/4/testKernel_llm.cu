#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void testKernel(int *s, const int *re) {
    __shared__ int temp[1];

    int i = threadIdx.x;

    // Use atomicMin to safely update temp[0] without race conditions
    if (re[i] > -1) {
        atomicMin(temp, re[i]);
    }

    __syncthreads();

    // Only one thread needs to write back the result
    if (i == 0) {
        *s = temp[0];
    }
}