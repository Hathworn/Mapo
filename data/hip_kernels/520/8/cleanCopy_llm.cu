#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cleanCopy(int *S, int *D){
    // Calculate the global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform copy operation only if within array bounds
    if(idx < N) // Assuming N is the size of the array
        D[idx] = S[idx];
}