#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void minValue(int *source, int *val) {
    __shared__ int temp[1];

    int currentValue = source[threadIdx.x];

    // Use atomic operation to update shared memory safely
    if (currentValue > -1) {
        atomicMin(temp, currentValue);
    }

    __syncthreads();

    // Only one thread writes back the minimum value
    if (threadIdx.x == 0) {
        atomicMin(val, temp[0]);
    }
}