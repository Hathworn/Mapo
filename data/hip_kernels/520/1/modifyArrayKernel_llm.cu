#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void modifyArrayKernel(int *val, int *arr) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < 6) { // Check i only once
        int current = arr[i];
        if (current > -1) {
            arr[i] = current - *val; // Minimize memory reads/writes
        }
    }
}