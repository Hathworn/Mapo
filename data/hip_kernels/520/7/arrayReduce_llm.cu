#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void arrayReduce(int *m, int *ms) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // Load ms[blockIdx.x] into shared memory for faster access
    __shared__ int shared_ms;
    if (threadIdx.x == 0) {
        shared_ms = ms[blockIdx.x];
    }
    __syncthreads();
    
    // Perform reduction only if condition is met
    if (m[id] > -1) {
        m[id] -= shared_ms; // Direct subtraction to optimize
    }
}