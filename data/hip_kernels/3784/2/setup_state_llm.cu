#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Pre-calculate the index using built-in variable for efficiency
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialize random state using unique thread index
    hiprand_init(seed, idx, 0, &state[idx]);
}