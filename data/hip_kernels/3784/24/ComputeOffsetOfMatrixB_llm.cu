#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Calculate global thread index
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (i < N) {
        // Compute and store the offset
        output[blockIdx.x * N + i] = -row_sum[blockIdx.x];
    }
}