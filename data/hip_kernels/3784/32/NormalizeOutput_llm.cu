#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Use shared memory for better memory access pattern and reduce latency
    extern __shared__ int64_t shared_mem[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Each thread loads elements into shared memory
    for (int i = idx; i < num_elements; i += stride) {
        shared_mem[threadIdx.x * 3] = batch_index;
        shared_mem[threadIdx.x * 3 + 1] = class_index;
        shared_mem[threadIdx.x * 3 + 2] = static_cast<int64_t>(original[i]);
        
        // Write shared memory back to global memory
        to_normalize[i * 3] = shared_mem[threadIdx.x * 3];
        to_normalize[i * 3 + 1] = shared_mem[threadIdx.x * 3 + 1];
        to_normalize[i * 3 + 2] = shared_mem[threadIdx.x * 3 + 2];
    }
}