#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixAB(const int32_t* __restrict__ row_sum, const int32_t* __restrict__ col_sum, int32_t* __restrict__ output, int32_t K_A_B, int32_t N) {
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index does not exceed bounds
    if (idx < N) {
        int32_t col_index = idx % blockDim.x;
        // Pre-compute constant row offset
        int32_t row_offset = K_A_B - row_sum[blockIdx.x];
        // Calculate output
        *(output + blockIdx.x * N + col_index) = row_offset - col_sum[col_index];
    }
}