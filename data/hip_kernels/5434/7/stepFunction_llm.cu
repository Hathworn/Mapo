#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stepFunction(const double dx, const int nx, double *u) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory for improved performance on each block
    __shared__ double shared_u[256]; // Adjust size based on block size
    for (int i = index; i < nx; i += stride) {
        
        double x = i * dx;
        // Run computation directly using x variable to avoid redundant calculations
        if (x >= 0.5 && x <= 1) {
            shared_u[threadIdx.x] = 2;
        } else {
            shared_u[threadIdx.x] = 1;
        }
        
        // Store back to global memory
        u[i] = shared_u[threadIdx.x];
    }
}