#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure not to access out of bounds
    if (i < size)
    {
        c[i] = a[i] + b[i];
    }
}