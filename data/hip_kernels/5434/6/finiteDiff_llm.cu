#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void finiteDiff(const int c, const double dt, const double dx, const int nt, const int nx, double *u, double *un) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    for (int t = 0; t < nt; t++) {
        if (index < nx) {  // Check to prevent out-of-bounds access
            // Use shared memory to minimize global memory access
            extern __shared__ double shared_un[];
            shared_un[index] = u[index];
            __syncthreads();

            if (index > 0) {
                u[index] = shared_un[index] - c * dt / dx * (shared_un[index] - shared_un[index - 1]);
            }
            __syncthreads();
        }
    }
}