#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xnor_gemm(unsigned int* A, unsigned int* B, float* C, int m, int n, int k) {

    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    float* Csub = &C[BLOCK_SIZE * k * blockRow + BLOCK_SIZE * blockCol];

    // Shared memory used to store Asub and Bsub respectively
    __shared__ unsigned int As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ unsigned int Bs[BLOCK_SIZE][BLOCK_SIZE];

    // Each thread computes one element of Csub by accumulating results into Cvalue
    unsigned int Cvalue = 0;

    // Loop over all the sub-matrices of A and B that are required to compute Csub
    for (int i = 0; i < (n / BLOCK_SIZE); ++i) {

        // Get sub-matrix Asub of A
        unsigned int* Asub = &A[BLOCK_SIZE * blockRow * n + BLOCK_SIZE * i];

        // Get sub-matrix Bsub of B
        unsigned int* Bsub = &B[BLOCK_SIZE * k * i + BLOCK_SIZE * blockCol];

        // Load Asub and Bsub from device memory to shared memory
        As[row][col] = Asub[row*n+col];
        Bs[row][col] = Bsub[row*k+col];

        // Synchronize to make sure the sub-matrices are loaded before starting the computation
        __syncthreads();

        // Optimize loop for better performance
        #pragma unroll // Unroll the inner loop for faster execution
        for (int j = 0; j < BLOCK_SIZE; ++j) {
            Cvalue += __popc(As[row][j] ^ Bs[j][col]);
        }

        // Synchronize to ensure completion of computation before loading new sub-matrices
        __syncthreads();
    }

    // Write Csub to device memory
    if (col + blockCol*BLOCK_SIZE < k && row + blockRow*BLOCK_SIZE < m) {
        Csub[row*k+col] = -(2*(float)Cvalue - 32*n);
    }
}