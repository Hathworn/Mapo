#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int concatenate(float* array) {
    unsigned int rvalue = 0;
    unsigned int sign;

    #pragma unroll // Unroll the loop for better performance
    for (int i = 0; i < 32; i++) {
        sign = (array[i] >= 0);
        rvalue = rvalue | (sign << i);
    }

    return rvalue;
}

__global__ void concatenate_cols_kernel(float *a, unsigned int *b, int m, int n) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (j < n) {
        // Use shared memory for better memory access performance
        __shared__ float array[32];

        for (int i = 0; i < m; i += 32) {
            #pragma unroll // Unroll the loop for better performance
            for (int k = 0; k < 32; k++) {
                array[k] = a[j + n * (i + k)];
            }
            b[j + n * i / 32] = concatenate(array);
        }
    }
}