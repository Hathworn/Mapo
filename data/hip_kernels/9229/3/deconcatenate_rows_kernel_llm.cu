#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ float* deconcatenate(unsigned int x)
{
    float * array = new float[32];

    for (int i = 0; i < 32; i++)
    {
        array[i] = (x & (1 << i)) >> i;
    }

    return array;
}

__device__ unsigned int concatenate(float* array)
{
    unsigned int rvalue = 0;
    unsigned int sign;

    for (int i = 0; i < 32; i++)
    {
        sign = (array[i] >= 0);
        rvalue = rvalue | (sign << i);
    }

    return rvalue;
}

__global__ void deconcatenate_rows_kernel(unsigned int *a, float *b, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Use thread index for parallelism
    if (idx < size / 32)
    {
        unsigned int val = a[idx];
        for (int k = 0; k < 32; k++)    // Inline deconcatenate for efficiency
        {
            b[idx * 32 + k] = (val & (1 << k)) >> k;
        }
    }
}