#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gemm(float* A, float* B, float* C, int m, int n, int k) {
    // Block row and column
    int blockRow = blockIdx.y;
    int blockCol = blockIdx.x;

    // Thread row and column within Csub
    int row = threadIdx.y;
    int col = threadIdx.x;

    // Each thread block computes one sub-matrix Csub of C
    float* Csub = &C[BLOCK_SIZE * k * blockRow + BLOCK_SIZE * blockCol];

    // Shared memory used to store Asub and Bsub respectively
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
    
    // Each thread computes one element of Csub by accumulating results into Cvalue
    float Cvalue = 0.0;

    // Loop over all the sub-matrices of A and B that are required to compute Csub
    for (int i = 0; i < (n / BLOCK_SIZE); ++i) {
        // Get sub-matrix Asub of A and Bsub of B
        float* Asub = &A[BLOCK_SIZE * blockRow * n + BLOCK_SIZE * i];
        float* Bsub = &B[BLOCK_SIZE * k * i + BLOCK_SIZE * blockCol];
        
        // Load Asub and Bsub from device memory to shared memory
        As[row][col] = Asub[row * n + col];
        Bs[row][col] = Bsub[row * k + col];

        // Synchronize to make sure the sub-matrices are loaded before starting the computation
        __syncthreads();

        // Unroll loop for better performance
        #pragma unroll
        for (int j = 0; j < BLOCK_SIZE; ++j) {
            Cvalue += As[row][j] * Bs[j][col];
        }

        // Synchronize to make sure that the preceding computation is done before loading two new sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write Csub to device memory
    if (col + blockCol * BLOCK_SIZE < k && row + blockRow * BLOCK_SIZE < m) {
        Csub[row * k + col] = Cvalue;
    }
}