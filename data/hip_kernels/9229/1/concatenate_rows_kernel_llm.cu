#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Concatenate function remains unchanged
__device__ unsigned int concatenate(float* array)
{
    unsigned int rvalue = 0;
    unsigned int sign;

    for (int i = 0; i < 32; i++)
    {
        sign = (array[i] >= 0);
        rvalue = rvalue | (sign << i);
    }

    return rvalue;
}

__global__ void concatenate_rows_kernel(float *a, unsigned int *b, int size)
{
    // Optimize index calculation with grid-stride loop for better utilization
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x)
    {
        b[i] = concatenate(&a[i * 32]);
    }
}