#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mapPrefixSumToPrisms(const unsigned numberOfPrisms, const unsigned raysPerSample, const unsigned reflectionSlices, const unsigned* raysPerPrism, const unsigned* prefixSum, unsigned* indicesOfPrisms, unsigned* numberOfReflections) {

    int id = threadIdx.x + (blockIdx.x * blockDim.x);

    // Return early if id exceeds bounds
    if (id >= numberOfPrisms * reflectionSlices) return;

    // Calculate indexes
    const unsigned count            = raysPerPrism[id];
    const unsigned startingPosition = prefixSum[id];
    const unsigned reflection_i     = id / numberOfPrisms;
    const unsigned prism_i          = id % numberOfPrisms;

    // Use parallel loop with stride
    for (unsigned i = startingPosition; i < startingPosition + count; ++i) {
        indicesOfPrisms[i] = prism_i;
        numberOfReflections[i] = reflection_i;
    }
}