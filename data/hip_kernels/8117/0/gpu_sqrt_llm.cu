#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

long N = 6400000000;
int doPrint = 0;

///////////////////////////////////////////////////////////////////////////////////////////////////////////
// HELPER CODE TO INITIALIZE, PRINT AND TIME
struct timeval start, end;

__global__ void gpu_sqrt(float* a, long N) {
    long element = blockIdx.x * blockDim.x + threadIdx.x;
    long stride = gridDim.x * blockDim.x; // Calculate stride for each block
    for (long i = element; i < N; i += stride) {
        a[i] = sqrt(a[i]); // Efficient global memory access with coalescing
    }
}