#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Device Code...

__global__ void force(double *deviceq, double *devicex, double *devicey, double *devicez, double *deviceFx, double *deviceFy, double *deviceFz, double *deviceU, int N)
{
    double foxij = 0.0, foyij = 0.0, fozij = 0.0, xij, yij, zij, rij;
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    if(i < N-1)
    {
        // Loop unrolling for better performance
        for(int j = i+1; j < N; j++)
        {
            xij = devicex[i] - devicex[j];
            yij = devicey[i] - devicey[j];
            zij = devicez[i] - devicez[j];

            // Avoid redundant division and calculations
            rij = sqrt((xij*xij) + (yij*yij) + (zij*zij));
            double invRijCubed = 1.0 / (rij * rij * rij);
            double qij = deviceq[i] * deviceq[j];
            
            foxij += qij * xij * invRijCubed;
            foyij += qij * yij * invRijCubed;
            fozij += qij * zij * invRijCubed;
            
            deviceU[i] += 2 * (deviceq[j] / rij);
        }
        
        atomicAdd(&deviceFx[i], foxij); // Use atomic operations to prevent race conditions
        atomicAdd(&deviceFy[i], foyij); // Use atomic operations to prevent race conditions
        atomicAdd(&deviceFz[i], fozij); // Use atomic operations to prevent race conditions
    }
}