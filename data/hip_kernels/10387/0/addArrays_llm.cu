#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void addArrays(int* a, int* b, int* c, int size)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within array bounds
    if (i < size) {
        // Perform addition
        c[i] = a[i] + b[i];
    }
}