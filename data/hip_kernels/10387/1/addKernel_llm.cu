#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate unique global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Perform addition only if within bounds
    if (i < blockDim.x * gridDim.x) {
        c[i] = a[i] + b[i];
    }
}