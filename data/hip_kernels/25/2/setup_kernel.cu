#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_kernel (hiprandState * state, unsigned long seed)
{
int id = blockIdx.x*NUM_BLOCKS + threadIdx.x;
hiprand_init ( seed, id, 0, &state[id] );
}