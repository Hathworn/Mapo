#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel(hiprandState *state, unsigned long seed)
{
    // Use grid dimensionality for better mapping of threads to state array
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialize random state for each thread
    hiprand_init(seed, id, 0, &state[id]);
}