#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void vcopyfrom(const int n, const int shift, const double *a, double *b)
{
    // Use faster memory access by calculating global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Improve parallelism with loop unrolling; Process multiple elements per thread
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x)
    {
        b[idx] = a[idx + shift];
    }
}