#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void tx1mx(const int lengthX, const double *t, const double *x, double *z)
{
    // Use shared memory for better memory coalescing
    extern __shared__ double sharedX[];
    
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < lengthX)
    {
        // Load data into shared memory
        sharedX[threadIdx.x] = x[i];
        __syncthreads();

        // Perform computation with shared memory
        double xi = sharedX[threadIdx.x];
        z[i] += t[i] * xi * (1.0 - xi);
    }
}