#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void xsigny_update(const int n, const double *a, double *b, double *c)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n)
    {
        // Simplify conditional check using conditional operator
        c[i] += (b[i] > 0) ? a[i] : (b[i] < 0) ? -a[i] : 0;
    }
}