#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void relu(const int n, const double *a, double *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Use ternary operator for conditional assignment
        b[i] = (a[i] > 0.0) ? a[i] : 0.0;
    }
}