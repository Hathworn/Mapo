#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// B=diagm(A)

extern "C"
{
}
__global__ void diagm_kernel(const int lengthA, const double *a, double *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride to handle elements beyond grid size
    for (int idx = i; idx < lengthA; idx += blockDim.x * gridDim.x)
    {
        b[idx + idx * lengthA] = a[idx];
    }
}