#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// B=diag(A)

extern "C"
{
}
__global__ void gfill(const int n, const double *a, double *c)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use conditional execution to reduce branching
    if (i < n)
    {
        c[i] = a[0];
    }
}