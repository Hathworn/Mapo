#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{
}

__global__ void vsquare(const double *a, double *c)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use conditional to prevent out-of-bounds memory access
    if (i < N)  // Replace N with the maximum index or size of the array
    {
        double v = a[i];
        c[i] = v * v;
    }
}