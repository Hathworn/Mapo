#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DYbinaryentropyXsigmoidY(const int lengthX, const double *x, const double *y, const double *t, double *z)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < lengthX)
    {
        // Precompute reused values for efficiency
        double exp_neg_y = exp(-y[i]);
        double sigmoid_y = 1.0 / (1.0 + exp_neg_y);
        
        // Perform the main computation
        z[i] += t[0] * (sigmoid_y - x[i]) / lengthX;
    }
}