#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to compute binary entropy
__global__ void binaryentropy(const int lengthX, const double *x, const double *y, double *z)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index

    if (i < lengthX) 
    {
        double xi = x[i]; // Cache x[i] in register
        double yi = y[i]; // Cache y[i] in register

        // Compute binary entropy
        z[i] = xi * log(xi / yi) + (1.0 - xi) * log((1.0 - xi) / (1.0 - yi));
    }
}