#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void serialsum(const int n, const double *x, double *y)
{
    // Using shared memory to improve access speed
    __shared__ double cache[256];

    int tid = threadIdx.x;
    double temp_sum = 0.0;

    // Accumulate partial sums
    for (int i = tid; i < n; i += blockDim.x)
    {
        temp_sum += x[i];
    }

    // Store the partial result in shared memory
    cache[tid] = temp_sum;
    __syncthreads();

    // Use binary reduction to sum up partial results
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        if (tid < stride)
        {
            cache[tid] += cache[tid + stride];
        }
        __syncthreads();
    }

    // Write the result from the first thread of the block
    if (tid == 0)
    {
        atomicAdd(y, cache[0]);
    }
}