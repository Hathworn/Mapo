#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// a simple CUDA kernel to element multiply two vectors C=alpha*A.*B

extern "C"   // ensure function name to be exactly "vmult!"
{
}

__global__ void vdivbang(const int lengthA, const double alpha, const double *a, const double *b, double *c)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    
    // Unrolling loop for coalesced global memory access and better occupancy
    int stride = gridDim.x * blockDim.x;
    while (i < lengthA) {
        c[i] = alpha * a[i] / b[i];
        i += stride;
    }
}