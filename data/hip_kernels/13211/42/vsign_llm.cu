#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vsign(const int n, const double *a, double *b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n)
    {
        double ai = a[i]; // Cache a[i] to reduce memory access
        if (ai < 0)
        {
            b[i] = -1.0;
        }
        else if (ai > 0)
        {
            b[i] = 1.0;
        }
        else
        {
            b[i] = 0.0;
        }
    }
}