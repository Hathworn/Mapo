#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access and kernel execution
extern "C"
{
}

__global__ void CalpahGaxpGy(const double alpha, const double *a, const double *b, double *c)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Avoid redundant memory access, process only within bounds
    if (i < N) // Assume N is the size of the array
    {
        double aVal = a[0]; // Load constant memory to register
        double bVal = b[i]; // Load once to register
        c[i] = alpha * aVal * bVal + c[i]; // Perform computation
    }
}