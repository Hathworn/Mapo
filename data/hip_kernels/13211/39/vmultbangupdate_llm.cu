#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vmultbangupdate(const int lengthA, const double alpha, const double *a, const double *b, double *c)
{
    // Calculate global thread index using a flat 1D grid mapping
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use strided loop to utilize parallelism more efficiently
    for (int idx = i; idx < lengthA; idx += blockDim.x * gridDim.x)
    {
        c[idx] += alpha * a[idx] * b[idx];
    }
}