#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// a simple CUDA kernel to add two vectors

extern "C" // ensure function name to be exactly "gaxpy2"
{
}

__global__ void CalpahGax(const double alpha, const double *a, const double *b, double *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Reorder for clarity
    // Check if index is within bounds to avoid out-of-bounds access
    if (i < N) 
    {
        c[i] = alpha * a[0] * b[i];
    }
}