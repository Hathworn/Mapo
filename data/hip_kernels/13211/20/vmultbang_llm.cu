#include "hip/hip_runtime.h"
```cuda
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void vmultbang(const int lengthA, const double alpha, const double *a, const double *b, double *c)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x; // Calculate the stride for each thread

    for (; i < lengthA; i += stride) // Use stride loop for increased efficiency
    {
        c[i] = alpha * a[i] * b[i];  // Perform the computation
    }
}