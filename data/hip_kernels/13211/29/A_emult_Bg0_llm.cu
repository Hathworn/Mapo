#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void A_emult_Bg0(const int n, const double *a, const double *b, double *c)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use unswitching to eliminate the redundant addition for b[i] <= 0
    if (i < n && b[i] > 0.0) {
        c[i] += a[i];
    }
}