#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void vcopyshift(const int n, const int shift, const double *a, double *b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global index
    if (i < n) // Ensure we do not access out of bounds
    {
        b[i + shift] = a[i]; // Perform the copy with shift
    }
}