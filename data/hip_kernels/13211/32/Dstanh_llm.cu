#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Unchanged kernel signature
extern "C" 
{
}

// Optimized kernel with loop unrolling and memory coalescing
__global__ void Dstanh(const int lengthX, const double sf, const double * __restrict__ gradc, const double * __restrict__ fc, double * __restrict__ gradn) 
{
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure proper memory alignment
    double local_sf = sf;

    // Unroll loop to process multiple elements per thread
    for (int i = idx; i < lengthX; i += blockDim.x * gridDim.x) {
        double fc_scaled = fc[i] / local_sf;
        gradn[i] += local_sf * gradc[i] * (1.0 - fc_scaled * fc_scaled);
    }
}