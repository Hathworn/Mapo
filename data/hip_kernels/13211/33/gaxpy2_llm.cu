#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void gaxpy2(const double *a, const double *b, double *c)
{
    int i = blockIdx.x * blockDim.x * blockDim.y * blockDim.z + threadIdx.z * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    // Use unique global thread index for better parallelization
    c[i] = a[0] * b[i] + c[i];  // REMEMBER ZERO INDEXING IN C LANGUAGE!!
}