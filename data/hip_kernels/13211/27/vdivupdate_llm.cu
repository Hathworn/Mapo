#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {}

__global__ void vdivupdate(const int lengthA, const double alpha, const double *a, const double *b, double *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop unrolling for better performance
    #pragma unroll
    for (; i < lengthA; i += gridDim.x * blockDim.x)
    {
        c[i] += alpha * a[i] / b[i];
    }
}