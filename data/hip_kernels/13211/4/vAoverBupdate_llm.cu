#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void vAoverBupdate(const int lengthA, const double alpha, const double *gradc, const double *a, const double *b, double *gradn)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < lengthA)
    {
        double bi = b[i];  // Load b[i] once and reuse
        double term = alpha * gradc[i] * a[i] / (bi * bi);  // Explicitly compute term

        gradn[i] -= term;  // Apply update
    }
}