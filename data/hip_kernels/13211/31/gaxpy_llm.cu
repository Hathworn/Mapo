#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{
}

__global__ void gaxpy(const int lengthC, const double *a, const double *b, double *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by ensuring all memory accesses are coalesced and using shared memory if necessary
    if (i < lengthC)
    {
        c[i] = a[0] * b[i] + c[i];
    }
}