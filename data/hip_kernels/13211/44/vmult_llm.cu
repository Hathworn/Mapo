#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vmult(const int lengthA, const double alpha, const double *a, const double *b, double *c)
{
    // Calculate the global index for the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the index within array bounds before computation
    if (i < lengthA)
    {
        // Perform the element-wise multiplication and scaling
        c[i] = alpha * a[i] * b[i];
    }
}