#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize the kernel function for better efficiency and readability
extern "C"
{
}

__global__ void binaryentropyXsigmoidY(const int lengthX, const double *x, const double *y, double *z)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < lengthX)
    {
        // Use registers to store recurring calculations
        double xi = x[i];
        double yi = y[i];
        
        // Simplify calculations by precomputing common expressions
        double log_xi = log(xi);
        double log_1_minus_xi = log(1.0 - xi);
        double exp_yi = exp(yi);
        
        // Compute the result using precomputed values
        z[i] = xi * log_xi + (1.0 - xi) * log_1_minus_xi - xi * yi + log(1.0 + exp_yi);
    }
}