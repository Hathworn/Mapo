#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// a simple CUDA kernel to element multiply two vectors C=alpha*A.*B

__global__ void gaxpy4(const int n, const double *a, const double *b, double *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use correct index calculation
    if (i < n) {
        c[i] = static_cast<double>(i); // Use static_cast for type casting
    }
}