#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void DXbinaryentropy(const int lengthX, const double *x, const double *y, const double *t, double *z)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure we do not perform calculations for threads beyond the array bounds
    if (i < lengthX)
    {
        // Precompute constant values and avoid repetitive function calls
        double xi = x[i];
        double yi = y[i];
        double log_term = log(xi * (1.0 - yi) / (yi * (1.0 - xi)));
        
        z[i] += t[0] * log_term / lengthX;
    }
}