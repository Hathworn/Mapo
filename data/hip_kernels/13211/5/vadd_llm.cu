#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void vadd(const int n, const double *a, const double *b, double *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride for better memory coalescing and load balancing
    for (; i < n; i += blockDim.x * gridDim.x)  
    {
        c[i] = a[i] + b[i];
    }
}