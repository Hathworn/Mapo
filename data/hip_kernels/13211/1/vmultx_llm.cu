#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use inline for small kernel for performance
__inline__ __device__ double multiply(const double alpha, const double a, const double b) {
    return alpha * a * b;
}

extern "C"   // ensure function name to be exactly "vmultx"
{
    // Launch kernel with a 1D grid and block structure
    __global__ void vmultx(const double alpha, const double *a, const double *b, double *c)
    {
        int i = threadIdx.x + blockIdx.x * blockDim.x;

        // Check bounds to prevent out-of-bounds access
        if (i < (blockDim.x * gridDim.x)) {
            c[i] = multiply(alpha, a[i], b[i]);
        }
    }
}