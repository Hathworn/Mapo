#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void vcopyfrom_update(const int n, const int shift, const double *a, double *b)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n)  // Boundary check
    {
        b[i] += __ldg(&a[i + shift]);  // Use __ldg for read-only data cache
    }
}