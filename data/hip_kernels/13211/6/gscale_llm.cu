#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void gscale(const int lengthB, const double *a, double *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use coalesced memory accesses by ensuring each thread accesses consecutive memory locations.
    if (i < lengthB)
    {
        double scale_factor = a[0]; // Store the constant factor in a local variable to avoid repeated global memory reads.
        b[i] *= scale_factor; // Optimize memory access by reading a[0] once.
    }
}