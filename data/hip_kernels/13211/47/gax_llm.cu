#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void gax(const int lengthC, const double *a, const double *b, double *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared variable for a[0] to reduce global memory access
    __shared__ double a_shared;
    if (threadIdx.x == 0) {
        a_shared = a[0];
    }
    __syncthreads();
    
    if (i < lengthC)
    {
        c[i] = a_shared * b[i];
    }
}