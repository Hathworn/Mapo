#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// filename: vsquare.cu
// a simple CUDA kernel to element multiply vector with itself

extern "C" // ensure function name to be exactly "vsquare"
{
}

__global__ void expkernel(const int lengthA, const double *a, double *b)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Unroll loops for better memory coalescing
    int stride = blockDim.x * gridDim.x;
    for (int index = i; index < lengthA; index += stride)
    {
        b[index] = exp(a[index]);
    }
}