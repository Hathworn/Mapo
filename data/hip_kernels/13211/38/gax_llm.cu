#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

// Kernel function with loop unrolling optimization
__global__ void gax(const int lengthC, const double *a, const double *b, double *c)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // Process two elements per iteration if possible, for better memory coalescing
    if (i < lengthC)
    {
        c[i] = a[0] * b[i];
    }
    if (i + blockDim.x * gridDim.x < lengthC)
    {
        c[i + blockDim.x * gridDim.x] = a[0] * b[i + blockDim.x * gridDim.x];
    }
}