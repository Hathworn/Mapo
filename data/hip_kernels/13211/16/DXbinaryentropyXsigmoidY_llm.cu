#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void DXbinaryentropyXsigmoidY(const int lengthX, const double *x,  const double *y, const double *t, double *z)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we are within bounds
    if (i < lengthX)
    {
        // Read x[i] once and store it in a register to reduce global memory access
        double xi = x[i];

        // Compute log term and subtraction in a single step
        double log_term = log(xi / (1.0 - xi));
        
        // Update z[i] using only necessary arithmetic operations
        z[i] -= t[0] * (y[i] - log_term) / lengthX;
    }
}