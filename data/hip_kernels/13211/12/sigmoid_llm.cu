#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"   // ensure function name to be exactly "gax"
{
}

__global__ void sigmoid(const int lengthA, const double *a, double *b)
{
    // Use a single load and computation to improve performance
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < lengthA)
    {
        double val = a[i];          // Cache a[i] in register
        val = exp(-val);            // Compute exp(-a[i])
        b[i] = 1.0 / (1.0 + val);   // Compute 1.0 / (1.0 + exp(-a[i]))
    }
}