#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to perform element-wise multiplication of a vector
extern "C" // ensure function name to be exactly "vmult!"
{
}

__global__ void diag_kernel(const int lengthA, const double *a, double *b)
{
    // Calculate the global index
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure index is within bounds and perform operation
    if (i < lengthA) {
        // Access diagonal element more efficiently
        int index = i * (lengthA + 1);
        b[i] = a[index];
    }
}