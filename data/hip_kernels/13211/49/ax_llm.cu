#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void ax(const int lengthC, const double a, const double *b, double *c)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  // Calculate total stride
    // Use stride to process multiple elements per thread
    for (; i < lengthC; i += stride)
    {
        c[i] = a * b[i];
    }
}