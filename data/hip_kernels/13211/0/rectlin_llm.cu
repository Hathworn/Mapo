#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
{
}

__global__ void rectlin(const int n, const double *a, double *b)
{
    // Use grid-stride loop for better parallelism
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < n; i += blockDim.x * gridDim.x) 
    {
        // Use ternary operator for clearer and compact code
        b[i] = (a[i] > 0.0) ? a[i] : 0.0;
    }
}