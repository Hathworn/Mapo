#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void stanh(const int lengthA, const double alpha, const double * __restrict__ a, double * __restrict__ b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = blockDim.x * gridDim.x;

    for (; i < lengthA; i += gridStride)  // Loop to handle larger arrays efficiently
    {
        b[i] = alpha * tanh(a[i]);  // Perform computation
    }
}