#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// filename: vmult!.cu
// a simple CUDA kernel to element multiply two vectors C=alpha*A.*B

extern "C" // ensure function name to be exactly "vmult!"
{
}

__global__ void vabs(const int n, const double *a, double *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop to improve memory coalescing
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x)
    {
        b[idx] = fabs(a[idx]);
    }
}