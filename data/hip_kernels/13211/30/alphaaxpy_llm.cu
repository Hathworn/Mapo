#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}
__global__ void alphaaxpy(const int lengthC, const double alpha, const double *a, const double *b, double *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < lengthC)
    {
        // Use registers to potentially lower memory load/store latencies
        double ai = a[0];
        double bi = b[i];
        double ci = c[i];
        
        // Perform computation
        c[i] = alpha * ai * bi + ci;
    }
}