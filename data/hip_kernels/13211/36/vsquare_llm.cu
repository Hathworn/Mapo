#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void vsquare(const double *a, double *c, int n)
{
    // Calculate global index
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // Check bounds to avoid accessing memory out of range
    if (i < n) {
        c[i] = a[i] * a[i];
    }
}