#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DmeanSquareLoss(const int lengthx, const double pref, const double *gradc, const double *x, const double *y, double *gradn)
{
    // Efficiently process multiple elements per thread for better utilization
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    
    for (int i = idx; i < lengthx; i += stride) {
        gradn[i] += pref * gradc[0] * (x[i] - y[i]);
    }
}