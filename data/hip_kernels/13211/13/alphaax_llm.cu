#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void alphaax(const int lengthC, const double alpha, const double *a, const double *b, double *c)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure i is within bounds
    if (i < lengthC)
    {
        // Preload a[0] for all threads once, if possible, before kernel launch
        double a0 = a[0];
        
        // Compute result
        c[i] = alpha * a0 * b[i];
    }
}