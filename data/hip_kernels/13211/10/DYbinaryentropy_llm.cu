#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void DYbinaryentropy(const int lengthX, const double *x, const double *y, const double *t, double *z)
{
    // Use shared memory cache for 't[0]' for better performance
    __shared__ double shared_t;
    if (threadIdx.x == 0) {
        shared_t = t[0];
    }
    __syncthreads();

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < lengthX)
    {
        // Optimize with registers: precompute expressions used multiple times
        double yi = y[i];
        double diff = y[i] - x[i];
        double denom = yi * (1.0 - yi);
        z[i] += shared_t * (diff / denom) / lengthX;
    }
}