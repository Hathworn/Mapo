#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
{
}

__global__ void gaxpy3(const int n, const double *a, const double *b, double *c)
{
    // Calculate the global index for each thread
    int i = blockIdx.x * blockDim.x + threadIdx.x 
            + (blockIdx.y * blockDim.y + threadIdx.y) * blockDim.x
            + (blockIdx.z * blockDim.z + threadIdx.z) * blockDim.x * blockDim.y;
    
    if (i < n) {
        // Efficient memory access and removal of unnecessary print statements
        c[i] = a[0] * b[i] + c[i];
    }
}