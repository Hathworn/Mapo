#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

struct compressed_sparse_column {
    int* data;
    int* row;
    int* column;
    int* index_column;
    int* index_row_start;
    int* index_row_end;
};

struct graph {
    compressed_sparse_column* dataset;
    bool* roots;
    bool* leaves;
    bool* singletons;
    int vertices;
    int edges;
};

__global__ void pre_post_order(int* depth, int* zeta, int* zeta_tilde, graph* dataset_graph) {
    extern __shared__ bool shared_memory[]; // Shared memory allocation
    int* pre = reinterpret_cast<int*>(shared_memory);
    int* post = pre + dataset_graph->vertices;
    bool* incoming_edges = reinterpret_cast<bool*>(post + dataset_graph->vertices);
    bool* q = reinterpret_cast<bool*>(incoming_edges + dataset_graph->edges);
    bool* p = reinterpret_cast<bool*>(q + dataset_graph->vertices);

    int tid = threadIdx.x;

    if (tid < dataset_graph->vertices) {
        pre[tid] = 0;
        post[tid] = 0;
        q[tid] = dataset_graph->roots[tid];
        p[tid] = false;
    }
    if (tid < dataset_graph->edges) {
        incoming_edges[tid] = false;
    }

    __syncthreads(); // Synchronize threads after initializing shared memory

    while (true) {
        bool global_check = false;

        for (int i = tid; i < dataset_graph->vertices; i += blockDim.x) {
            if (q[i]) {
                int pre_node = pre[i];
                int post_node = post[i];

                for (int j = dataset_graph->dataset->index_column[i];
                     dataset_graph->dataset->column[j] == i; j++) {
                    int neighbor_vertex = dataset_graph->dataset->row[j];

                    pre[neighbor_vertex] = pre_node + zeta_tilde[neighbor_vertex];
                    post[neighbor_vertex] = post_node + zeta_tilde[neighbor_vertex];
                    incoming_edges[j] = true;

                    bool flag = true;
                    for (int k = 0; k < dataset_graph->edges; k++) {
                        if (dataset_graph->dataset->row[k] == neighbor_vertex && !incoming_edges[k]) {
                            flag = false;
                            break;
                        }
                    }
                    if (flag) {
                        global_check = true;
                        p[neighbor_vertex] = true;
                    }
                }
                pre[i] = pre_node + depth[i];
                post[i] = post_node + (zeta[i] - 1);
            }
        }

        __syncthreads(); // Synchronize threads before updating `q`

        if (tid < dataset_graph->vertices) {
            q[tid] = p[tid];
            p[tid] = false;
        }

        __syncthreads(); // Synchronize updated `q` across threads

        if (!global_check) {
            break;
        }
    }
}