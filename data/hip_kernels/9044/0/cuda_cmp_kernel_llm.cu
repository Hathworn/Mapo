#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function
__global__ void cuda_cmp_kernel(std::size_t n, int* aptr, int* bptr, int* rptr) {
    // Compute global thread index
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Initialize shared memory to store comparison
    __shared__ int shared_cmp;
    if (threadIdx.x == 0) {
        shared_cmp = 0;
    }
    __syncthreads();

    // Perform comparison and store result in shared memory
    if (i < n && aptr[i] < bptr[i]) {
        atomicAdd(&shared_cmp, 1);
    }
    __syncthreads();

    // Write global result from shared memory
    if (threadIdx.x == 0 && shared_cmp > 0) {
        *rptr = 1;
    }
}