#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

static char* program_name;

// Usage
__global__ void jacobiOptimizedOnDevice(float* x_next, float* A, float* x_now, float* b, int Ni, int Nj)
{
  // Use shared memory to reduce global memory access
  __shared__ float s_x_now[1024];  // assuming Nj <= 1024, adjust size as needed
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // Load x_now into shared memory
  if (threadIdx.x < Nj) {
    s_x_now[threadIdx.x] = x_now[threadIdx.x];
  }
  __syncthreads();  // synchronize to ensure all threads have loaded x_now
  
  if (idx < Ni)
  {
    float sigma = 0.0;
    int idx_Ai = idx * Nj;

    // Iterate using shared memory for x_now values
    for (int j = 0; j < Nj; j++) {
      if (idx != j) {
        sigma += A[idx_Ai + j] * s_x_now[j];
      }
    }

    // Calculate the next value of x
    x_next[idx] = (b[idx] - sigma) / A[idx_Ai + idx];
  }
}