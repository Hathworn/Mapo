#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

static char* program_name;

// Optimized kernel using shared memory
__global__ void jacobiOnDevice(float* x_next, float* A, float* x_now, float* b, int Ni, int Nj)
{
    extern __shared__ float shared_x_now[]; // Allocate shared memory for x_now

    int idx = threadIdx.x;
    if (idx < Nj) {
        shared_x_now[idx] = x_now[idx]; // Load x_now into shared memory
    }
    __syncthreads(); // Ensure all threads have loaded x_now to shared memory

    if (idx < Nj) {
        float sigma = 0.0;
        for (int j = 0; j < Nj; j++) {
            if (idx != j) {
                sigma += A[idx * Nj + j] * shared_x_now[j]; // Use shared memory
            }
        }
        x_next[idx] = (b[idx] - sigma) / A[idx * Nj + idx];
    }
}