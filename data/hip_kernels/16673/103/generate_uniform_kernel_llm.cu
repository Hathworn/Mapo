#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_uniform_kernel(float *generated_out, hiprandState_t *state)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index

    if(idx < gridDim.x * blockDim.x) // Ensure idx is within bounds
    {
        generated_out[idx] = hiprand_uniform(&state[idx]); // Generate a uniform random number
    }
}