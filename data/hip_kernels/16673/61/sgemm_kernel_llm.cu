#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Use shared memory for tiles of A and B
    __shared__ float Asub[32][32];
    __shared__ float Bsub[32][32];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.f;

    // Loop over A and B tiles
    for (int tile = 0; tile < (K-1)/32+1; ++tile) {
        if (tile * 32 + threadIdx.x < K && row < N)
            Asub[threadIdx.y][threadIdx.x] = A[row * K + tile * 32 + threadIdx.x];
        else
            Asub[threadIdx.y][threadIdx.x] = 0.f;

        if (tile * 32 + threadIdx.y < K && col < M)
            Bsub[threadIdx.y][threadIdx.x] = B[(tile * 32 + threadIdx.y) * M + col];
        else
            Bsub[threadIdx.y][threadIdx.x] = 0.f;

        __syncthreads();

        // Multiply the tiles
        for (int i = 0; i < 32; ++i)
            sum += Asub[threadIdx.y][i] * Bsub[i][threadIdx.x];

        __syncthreads();
    }

    if(row < N && col < M)
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
}