#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd_kernel(float *c, const float* a, const float* b)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop for better performance
    #pragma unroll 5
    for (int i = 0; i < 500; i++)
    {
        c[idx] = a[idx] + b[idx];
    }
}