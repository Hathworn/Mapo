#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < N && col < M) { // Boundary check to ensure row and col are within limits
        float sum = 0.f;
        for (int i = 0; i < K; ++i) {
            sum += A[row * K + i] * B[i * M + col]; // Corrected B matrix index
        }
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}