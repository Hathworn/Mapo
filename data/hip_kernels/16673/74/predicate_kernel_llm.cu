#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void predicate_kernel(float *d_predicates, float *d_input, int length)
{
    // Calculate global thread index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure idx is within bounds using a grid-stride loop for better parallelism
    for (int i = idx; i < length; i += gridDim.x * blockDim.x) {
        d_predicates[i] = d_input[i] > FLT_ZERO;
    }
}