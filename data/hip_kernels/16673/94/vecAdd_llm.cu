#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID
    if (i < len) {
        out[i] = in1[i] + in2[i]; // Perform vector addition if within bounds
    }
}