#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd_kernel(float *c, const float* a, const float* b)
{
    // Calculate global index for each thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid unnecessary repeated calculations
    float sum = a[idx] + b[idx];
    for (int i = 0; i < 500; i++)
    {
        c[idx] = sum;
    }
}