#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction_kernel_2(float *g_out, float *g_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float s_data[];

    // Load data into shared memory with bounds checking
    s_data[threadIdx.x] = (idx_x < size) ? g_in[idx_x] : 0.f;

    __syncthreads();

    // Perform reduction using a more efficient loop
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        // Check and add in the same step
        if (threadIdx.x < stride)
        {
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Only the first thread writes back to global memory
    if (threadIdx.x == 0)
    {
        g_out[blockIdx.x] = s_data[0];
    }
}