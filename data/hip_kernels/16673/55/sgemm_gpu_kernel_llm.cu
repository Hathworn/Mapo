#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_gpu_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Using shared memory for tiles
    __shared__ float tileA[32][32];
    __shared__ float tileB[32][32];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int col = blockIdx.x * blockDim.x + tx;
    int row = blockIdx.y * blockDim.y + ty;

    float sum = 0.0f;

    for (int t = 0; t < (K + 31) / 32; ++t)
    {
        // Load tiles into shared memory
        if (row < N && t * 32 + tx < K)
            tileA[ty][tx] = A[row * K + t * 32 + tx];
        else
            tileA[ty][tx] = 0.0f;

        if (col < M && t * 32 + ty < K)
            tileB[ty][tx] = B[(t * 32 + ty) * M + col];
        else
            tileB[ty][tx] = 0.0f;

        __syncthreads();

        // Compute partial product for this tile
        for (int i = 0; i < 32; ++i)
        {
            sum += tileA[ty][i] * tileB[i][tx];
        }

        __syncthreads();
    }

    if (row < N && col < M)
    {
        // Write result back to global memory
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}