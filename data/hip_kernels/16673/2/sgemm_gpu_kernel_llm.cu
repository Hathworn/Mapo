#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_gpu_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Calculate global row and column indices
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure valid row and column indices
    if (row < N && col < M) {
        float sum = 0.f;

        // Use shared memory for better performance
        extern __shared__ float shmem[];
        float* As = shmem;
        float* Bs = shmem + blockDim.y;

        for (int i = 0; i < K; ++i) {
            As[threadIdx.y] = A[row * K + i];
            Bs[threadIdx.x] = B[i * M + col];
            __syncthreads();

            sum += As[threadIdx.y] * Bs[threadIdx.x];
            __syncthreads();
        }

        // Compute final output with alpha and beta scaling
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}