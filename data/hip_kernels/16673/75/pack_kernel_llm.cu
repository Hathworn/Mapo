#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pack_kernel(float *d_output, float *d_input, float *d_predicates, float *d_scanned, int length)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < length)
    {
        // Check predicate and gather in one operation
        if (d_predicates[idx] != 0.f)
        {
            int address = __float_as_int(d_scanned[idx]) - 1;
            d_output[address] = d_input[idx];
        }
    }
}