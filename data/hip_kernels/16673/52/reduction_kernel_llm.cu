#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction_kernel(float* d_out, float* d_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float s_data[];

    // Load input into shared memory, ensuring no out-of-bounds access
    s_data[threadIdx.x] = (idx_x < size) ? d_in[idx_x] : 0.f;

    __syncthreads();

    // Perform reduction using a more efficient loop structure
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
    {
        // Reduce within block using warp-level optimizations
        if (threadIdx.x < stride)
        {
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        // Write result for current block to global memory
        d_out[blockIdx.x] = s_data[0];
    }
}