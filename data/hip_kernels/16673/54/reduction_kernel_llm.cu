#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction_kernel(float *g_out, float *g_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ float s_data[];

    // Initialize shared memory with zero
    s_data[threadIdx.x] = 0.0f;

    // Accumulate input with unrolled grid-stride loop
    for (int i = idx_x; i < size; i += blockDim.x * gridDim.x)
    {
        s_data[threadIdx.x] += g_in[i];
    }

    __syncthreads();

    // Perform tree-based reduction
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride)
        {
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0)
    {
        g_out[blockIdx.x] = s_data[0];
    }
}