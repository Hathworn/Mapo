#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction_kernel(float* d_out, float* d_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ float s_data[];

    // Use conditional assignment to avoid bank conflicts
    s_data[threadIdx.x] = (idx_x < size) ? d_in[idx_x] : 0.f;
    __syncthreads();

    // Unrolling loop for efficiency
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride) // Only necessary threads perform work
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        
        __syncthreads(); // synchronize to ensure all operations are complete
    }

    // Thread 0 writes the result for this block to global memory
    if (threadIdx.x == 0)
        d_out[blockIdx.x] = s_data[0];
}