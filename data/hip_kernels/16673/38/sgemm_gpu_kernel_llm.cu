#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_gpu_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < N && col < M) {  // Boundary check to avoid out-of-bound memory access
        float sum = 0.f;

        // Loop unrolling for better performance
        #pragma unroll
        for (int i = 0; i < K; ++i) {
            sum += A[row * K + i] * B[i * M + col];  // Corrected indexing for B
        }

        C[row * M + col] = alpha * sum + beta * C[row * M + col]; // Storing the result
    }
}