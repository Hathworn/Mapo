#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan_v2_kernel(float *d_output, float *d_input, int length)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int tid = threadIdx.x;

    extern __shared__ float s_buffer[];
    s_buffer[tid] = d_input[idx];
    s_buffer[tid + BLOCK_DIM] = d_input[idx + BLOCK_DIM];

    int offset = 1;

    // Forward pass
    for (; offset < length; offset <<= 1) 
    {
        __syncthreads();

        int idx_a = offset * (2 * tid + 1) - 1;
        int idx_b = offset * (2 * tid + 2) - 1;

        // Prevent unnecessary divergence
        if (idx_b < 2 * BLOCK_DIM) 
        {
            if (idx_a >= 0) 
            {
                s_buffer[idx_b] += s_buffer[idx_a];
            }
        }
    }

    offset >>= 1;

    // Backward pass
    for (; offset > 0; offset >>= 1) 
    {
        __syncthreads();

        int idx_a = offset * (2 * tid + 2) - 1;
        int idx_b = offset * (2 * tid + 3) - 1;

        // Prevent unnecessary divergence
        if (idx_b < 2 * BLOCK_DIM) 
        {
            if (idx_a >= 0)
            {
                s_buffer[idx_b] += s_buffer[idx_a];
            }
        }
    }
    __syncthreads();

    // Write back to global memory
    d_output[idx] = s_buffer[tid];
    d_output[idx + BLOCK_DIM] = s_buffer[tid + BLOCK_DIM];
}