#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel_A(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Compute global thread row and column indices
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds to avoid illegal memory accesses
    if (row < N && col < M) {
        float sum = 0.f;
        // Unroll the loop for improved performance
        for (int i = 0; i < K; ++i) {
            sum += A[row * K + i] * B[i * K + col];
        }
        // Perform and store the result in C
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}