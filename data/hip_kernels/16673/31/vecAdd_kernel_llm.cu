#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd_kernel(float *c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilize shared memory to reduce global memory access
    __shared__ float shared_a[blockDim.x];
    __shared__ float shared_b[blockDim.x];
    
    // Load data into shared memory
    shared_a[threadIdx.x] = a[idx];
    shared_b[threadIdx.x] = b[idx];
    
    // Synchronize threads to ensure shared memory is loaded
    __syncthreads();
    
    // Perform computation using shared memory
    for (int i = 0; i < 500; i++) {
        // Accumulate in shared memory to avoid repeated global memory writes
        shared_a[threadIdx.x] += shared_b[threadIdx.x];
    }
    
    // Write result from shared memory to global memory
    c[idx] = shared_a[threadIdx.x];
}