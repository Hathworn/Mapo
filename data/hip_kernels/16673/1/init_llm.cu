#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init(int n, float *x, float *y) {
    // Using shared memory for frequently accessed data
    __shared__ size_t shared_warp_total;
    
    int lane_id = threadIdx.x & 31;
    size_t warp_id = (threadIdx.x + blockIdx.x * blockDim.x) >> 5;
    size_t warps_per_grid = (blockDim.x * gridDim.x) >> 5;
    
    // Calculate total warps once and store in shared memory
    if (threadIdx.x == 0) {
        shared_warp_total = ((sizeof(float) * n) + STRIDE_64K - 1) / STRIDE_64K;
    }
    __syncthreads();

    // Load shared memory to local for use in loop
    size_t warp_total = shared_warp_total;

    for (; warp_id < warp_total; warp_id += warps_per_grid) {
        #pragma unroll
        for (int rep = 0; rep < STRIDE_64K / sizeof(float) / 32; rep++) {
            size_t ind = warp_id * STRIDE_64K / sizeof(float) + rep * 32 + lane_id;
            if (ind < n) {
                x[ind] = 1.0f;
                y[ind] = 2.0f;
            }
        }
    }
}
```
