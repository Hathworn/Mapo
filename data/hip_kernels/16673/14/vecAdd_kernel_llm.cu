#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd_kernel(float *c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop unrolling for optimization
    #pragma unroll 5
    for (int i = 0; i < 500; i++)
    {
        c[idx] = a[idx] + b[idx];
    }
}