#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution_kernel_v1(float *d_output, float *d_input, float *d_filter, int num_row, int num_col, int filter_size)
{
    // Use shared memory for filter
    extern __shared__ float shared_filter[];

    int idx_x = blockDim.x * blockIdx.x + threadIdx.x;
    int idx_y = blockDim.y * blockIdx.y + threadIdx.y;

    // Load filter into shared memory
    int filter_index = threadIdx.y * blockDim.x + threadIdx.x;
    if (filter_index < filter_size * filter_size) {
        shared_filter[filter_index] = d_filter[filter_index];
    }
    __syncthreads(); // Ensure all threads have loaded the filter

    float result = 0.f;

    // Loop unrolling for filter rows and columns
    for (int filter_row = -filter_size / 2; filter_row <= filter_size / 2; ++filter_row)
    {
        for (int filter_col = -filter_size / 2; filter_col <= filter_size / 2; ++filter_col)
        {
            // Find the global position to apply the given filter
            int image_row = idx_y + filter_row;
            int image_col = idx_x + filter_col;

            float image_value = (image_row >= 0 && image_row < num_row && image_col >= 0 && image_col < num_col) ?
                d_input[image_row * num_col + image_col] : 0.f;
            
            // Access shared memory for filter value
            float filter_value = shared_filter[(filter_row + filter_size / 2) * filter_size + filter_col + filter_size / 2];

            result += image_value * filter_value;
        }
    }

    d_output[idx_y * num_col + idx_x] = result;
}