#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized HIP kernel to add elements of two arrays
__global__ void init(int n, float *x, float *y) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) { // Ensure work is not done beyond necessary
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
}