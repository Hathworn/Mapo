#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void createResizedImage(unsigned char *imageScaledData, int scaled_width, float scale_factor, hipTextureObject_t texObj) {
    // Calculate thread index in a coalesced manner for better memory access pattern
    const unsigned int tidX = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int tidY = blockIdx.y*blockDim.y + threadIdx.y;
    const unsigned int index = tidY * scaled_width + tidX;

    // Ensure that the thread index is within the valid range
    if (tidX < scaled_width && tidY < scaled_width) {
        // Optimized texture fetching; coalesced memory access
        imageScaledData[index] = tex2D<unsigned char>(texObj, tidX * scale_factor, tidY * scale_factor);
    }
}