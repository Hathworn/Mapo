#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction_kernel_1(float* g_out, float* g_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ float s_data[];

    // Load input elements into shared memory
    s_data[threadIdx.x] = (idx_x < size) ? g_in[idx_x] : 0.f;

    __syncthreads();

    // Optimize reduction using unrolling and warp-level primitives
    // Unroll the loop and use warp reduce
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride)
        {
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }
        __syncthreads();  // Ensure all threads have completed warp reduction
    }

    // Store the result from each block in the global memory
    if (threadIdx.x == 0)
    {
        g_out[blockIdx.x] = s_data[0];
    }
}