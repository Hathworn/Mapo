#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sgemm_kernel_v2(const float *A, const float *B, float *C, int M, int N, int K, float alpha, float beta)
{
    int bid_x = blockIdx.x * blockDim.x;
    int bid_y = blockIdx.y * blockDim.y;
    int tid_x = threadIdx.x;
    int tid_y = threadIdx.y;

    float element_c = 0.f;
    __shared__ float s_tile_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float s_tile_B[BLOCK_DIM][BLOCK_DIM];

    // Process tiles
    for (int k = 0; k < K; k += BLOCK_DIM)
    {
        // Load A and B matrices into shared memory efficiently
        s_tile_A[tid_y][tid_x] = (bid_y + tid_y) < M && (tid_x + k) < K ? A[(bid_y + tid_y) * K + tid_x + k] : 0;
        s_tile_B[tid_y][tid_x] = (tid_y + k) < K && (bid_x + tid_x) < N ? B[(k + tid_y) * N + bid_x + tid_x] : 0;

        __syncthreads();

        // Cache-friendly multiplication
        for (int e = 0; e < BLOCK_DIM; e++)
            element_c += s_tile_A[tid_y][e] * s_tile_B[e][tid_x];

        __syncthreads();
    }

    // Validate indices before writing result
    if ((bid_y + tid_y) < M && (bid_x + tid_x) < N)
    {
        C[(bid_y + tid_y) * N + (bid_x + tid_x)] = alpha * element_c + beta * C[(bid_y + tid_y) * N + (bid_x + tid_x)];
    }
}