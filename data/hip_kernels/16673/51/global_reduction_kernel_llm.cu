#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_reduction_kernel(float *data_out, float *data_in, int stride, int size)
{
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to minimize global memory access
    extern __shared__ float sdata[];
    
    if (idx_x < size) {
        sdata[threadIdx.x] = data_in[idx_x];
    } else {
        sdata[threadIdx.x] = 0.0f;
    }

    __syncthreads();

    if (idx_x + stride < size) {
        data_out[idx_x] = sdata[threadIdx.x] + sdata[threadIdx.x + stride];
    }
}