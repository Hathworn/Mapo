#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void print_from_gpu(void) {
    // Use block and thread id for unique message
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello World! from thread %d \
From device\n", tid);
}