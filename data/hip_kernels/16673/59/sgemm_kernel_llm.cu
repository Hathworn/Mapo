#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sgemm_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Calculate the column index
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the row index
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < N && col < M) // Ensure within bounds
    {
        // Use shared memory for block of B
        __shared__ float shared_B[BLOCK_SIZE][BLOCK_SIZE];
        float sum = 0.f;

        for (int tile = 0; tile < (K + BLOCK_SIZE - 1) / BLOCK_SIZE; ++tile) 
        {
            // Load A and B blocks into registers and shared memory
            float a_element = 0.f;
            if (tile * BLOCK_SIZE + threadIdx.x < K) 
            {
                a_element = A[row * K + tile * BLOCK_SIZE + threadIdx.x];
            }
            if (tile * BLOCK_SIZE + threadIdx.y < K && col < M) 
            {
                shared_B[threadIdx.y][threadIdx.x] = B[(tile * BLOCK_SIZE + threadIdx.y) * M + col];
            }
            else
            {
                shared_B[threadIdx.y][threadIdx.x] = 0.f;
            }
            __syncthreads(); // Ensure all elements are loaded before computation

            // Matrix multiplication using sub-matrices
            for (int i = 0; i < BLOCK_SIZE; ++i) 
            {
                sum += a_element * shared_B[i][threadIdx.x];
            }
            __syncthreads(); // Ensure computation is finished before loading next tiles
        }

        C[row * M + col] = alpha * sum + beta * C[row * M + col]; // Compute final sum
    }
}