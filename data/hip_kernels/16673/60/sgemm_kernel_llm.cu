#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if row and col are within matrix bounds
    if (row < N && col < M) {
        float sum = 0.f;
        // Improve memory access by handling coalesced reads
        for (int i = 0; i < K; ++i) {
            sum += A[row * K + i] * B[i * M + col];
        }
        // Efficiently compute the output element
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}