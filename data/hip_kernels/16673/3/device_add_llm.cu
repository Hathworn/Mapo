#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_add(int *a, int *b, int *c) {
    // Calculate global index
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Check if index is within bounds
    if(index < N) {
        c[index] = a[index] + b[index];
    }
}
```
Note: Make sure `N` is defined elsewhere in your code to specify the size of arrays.