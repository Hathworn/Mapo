#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void iterative_saxpy_kernel(float *y, const float* x, const float alpha, const float beta, int n_loop)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread index is within array bounds
    if (idx < n_loop) {
        float xi = x[idx];
        #pragma unroll // Suggest loop unrolling for performance
        for (int i = 0; i < n_loop; i++)
            y[idx] = alpha * xi + beta;
    }
}