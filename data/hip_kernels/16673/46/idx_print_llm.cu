#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void idx_print()
{
    // Calculate global thread index in a more concise way
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use bitwise operations to compute warp and lane indices
    int warp_idx = threadIdx.x / warpSize;
    int lane_idx = threadIdx.x & (warpSize - 1);

    // Optimize condition for printing using bitwise operations
    if ((lane_idx & ((warpSize >> 1) - 1)) == 0)
        // Printing the indices
        printf(" %5d\t%5d\t %2d\t%2d\n", idx, blockIdx.x, warp_idx, lane_idx);
}