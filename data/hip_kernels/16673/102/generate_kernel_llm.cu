#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_kernel(unsigned int *generated_out, hiprandState_t *state)
{
    // Use variables to cache the result of calculations involving blockIdx and threadIdx
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Directly assign the result of hiprand to reduce memory access
    generated_out[idx] = hiprand(&state[idx]);
}