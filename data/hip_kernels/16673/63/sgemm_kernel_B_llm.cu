#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel_B(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Use shared memory for faster access
    __shared__ float shared_A[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float shared_B[BLOCK_SIZE][BLOCK_SIZE];

    float sum = 0.f;

    // Loop over tiles
    for (int tileIdx = 0; tileIdx < (K + BLOCK_SIZE - 1) / BLOCK_SIZE; ++tileIdx)
    {
        // Load data into shared memory
        if (row < N && tileIdx * BLOCK_SIZE + threadIdx.x < K)
            shared_A[threadIdx.y][threadIdx.x] = A[row * K + tileIdx * BLOCK_SIZE + threadIdx.x];
        else
            shared_A[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < M && tileIdx * BLOCK_SIZE + threadIdx.y < K)
            shared_B[threadIdx.y][threadIdx.x] = B[(tileIdx * BLOCK_SIZE + threadIdx.y) * M + col];
        else
            shared_B[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        // Perform partial matrix multiplication
        for (int i = 0; i < BLOCK_SIZE; ++i)
            sum += shared_A[threadIdx.y][i] * shared_B[i][threadIdx.x];

        __syncthreads();
    }

    // Write final result into global memory
    if (row < N && col < M)
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
}