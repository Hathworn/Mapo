#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd_kernel(float *c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to optimize repeated memory access
    __shared__ float sharedA[500];
    __shared__ float sharedB[500];

    // Load data into shared memory
    if (idx < 500) {
        sharedA[idx] = a[idx];
        sharedB[idx] = b[idx];
    }
    __syncthreads();

    // Perform computation using shared memory
    if (idx < 500) {
        c[idx] = sharedA[idx] + sharedB[idx];
    }
}