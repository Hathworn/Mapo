#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction_kernel(float *g_out, float *g_in, unsigned int size)
{
    unsigned int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ float s_data[];

    // Initialize input array in shared memory to zero
    float sum = 0.f;

    // Grid-stride loop to accumulate inputs
    for (unsigned int i = idx_x; i < size; i += blockDim.x * gridDim.x) {
        sum += g_in[i];
    }

    s_data[threadIdx.x] = sum; // Store partial sum in shared memory

    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        // Only active threads perform the reduction
        if (threadIdx.x < stride) {
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        g_out[blockIdx.x] = s_data[0];
    }
}