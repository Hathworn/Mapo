#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel_B(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Shared memory for tiles
    __shared__ float As[16][16];
    __shared__ float Bs[16][16];

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0.f;
    
    // Loop over tiles
    for (int t = 0; t < (K + 15) / 16; t++) {
        // Load tiles into shared memory
        if(t * 16 + threadIdx.x < K && row < N)
            As[threadIdx.y][threadIdx.x] = A[row * K + t * 16 + threadIdx.x];
        else
            As[threadIdx.y][threadIdx.x] = 0.0f;

        if(t * 16 + threadIdx.y < K && col < M)
            Bs[threadIdx.y][threadIdx.x] = B[(t * 16 + threadIdx.y) * M + col];
        else
            Bs[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();  // Wait for all threads to load the tiles

        // Multiply and accumulate
        for (int i = 0; i < 16; ++i)
            sum += As[threadIdx.y][i] * Bs[i][threadIdx.x];

        __syncthreads();  // Ensure all threads complete calculations
    }

    if(row < N && col < M)
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
}