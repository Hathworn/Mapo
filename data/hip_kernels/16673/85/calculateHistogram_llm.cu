#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculateHistogram(unsigned int *imageHistogram, unsigned int width, unsigned int height, hipTextureObject_t texObj)
{
    const unsigned int tidX = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int tidY = blockIdx.y*blockDim.y + threadIdx.y;

    const unsigned int localId = threadIdx.y*blockDim.x + threadIdx.x;
    const unsigned int histStartIndex = (blockIdx.y*gridDim.x + blockIdx.x) * 256;

    __shared__ unsigned int histo_private[256];

    // Initialize shared histogram to zero using cooperative threads
    if (localId < 256)
    {
        histo_private[localId] = 0;
    }
    __syncthreads();

    // Check if within image bounds before accessing texture
    if (tidX < width && tidY < height)
    {
        // Efficiently use texture with integer indices
        unsigned char imageData = tex2D<unsigned char>(texObj, (float)(tidX), (float)(tidY));
        atomicAdd(&histo_private[imageData], 1);
    }
    __syncthreads();

    // Write back to global memory
    if (localId < 256)
    {
        imageHistogram[histStartIndex + localId] = histo_private[localId];
    }
}