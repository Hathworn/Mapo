#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_add(int *a, int *b, int *c) {
    // Calculate the global index for the current thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  
    // Perform the addition for each thread
    c[idx] = a[idx] + b[idx];
}