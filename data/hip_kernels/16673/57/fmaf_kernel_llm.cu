#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize stride calculation and loop condition for better memory access pattern and performance
__global__ void fmaf_kernel(float *d_x, float *d_y, float *d_z, int size)
{
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to minimize global memory access if possible
    __shared__ float shared_x[blockDim.x];
    __shared__ float shared_y[blockDim.x];
    
    for (int i = idx_x; i < size; i += blockDim.x * gridDim.x) {
        shared_x[threadIdx.x] = d_x[i]; // Load to shared memory
        shared_y[threadIdx.x] = d_y[i]; // Load to shared memory
        __syncthreads(); // Synchronize to ensure memory consistency
        
        // Perform fmaf operation using values in shared memory
        d_z[i] = fmaf(shared_x[threadIdx.x], shared_y[threadIdx.x], 0.f);

        __syncthreads(); // Synchronize before next iteration
    }
}