#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_add(int *a, int *b, int *c) {
    // Compute the global index once and reuse it
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Optimize memory access by using shared memory
    __shared__ int temp_a[1024];
    __shared__ int temp_b[1024];

    // Load data into shared memory
    temp_a[threadIdx.x] = a[index];
    temp_b[threadIdx.x] = b[index];
  
    // Ensure all threads have loaded their data
    __syncthreads();

    // Perform addition using shared memory
    c[index] = temp_a[threadIdx.x] + temp_b[threadIdx.x];
}