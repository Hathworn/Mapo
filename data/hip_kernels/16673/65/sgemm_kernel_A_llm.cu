#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel_A(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Use shared memory to optimize memory access
    extern __shared__ float shared_mem[];
    float* shared_A = shared_mem;
    float* shared_B = shared_mem + blockDim.y * K;

    // Calculate thread indices
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0.f;
    for (int i = 0; i < K; i += blockDim.y) {
        // Load A and B into shared memory
        if (row < N && (i + threadIdx.y) < K) {
            shared_A[threadIdx.y * K + i + threadIdx.x] = A[row * K + i + threadIdx.x];
        }
        if (col < M && (i + threadIdx.y) < K) {
            shared_B[threadIdx.y * K + i + threadIdx.x] = B[(i + threadIdx.y) * M + col];
        }
        
        // Synchronize after loading to shared memory
        __syncthreads();

        // Compute partial sum using shared memory
        for (int j = 0; j < blockDim.y; ++j) {
            sum += shared_A[threadIdx.y * K + j] * shared_B[j * M + threadIdx.x];
        }

        // Synchronize before next iteration
        __syncthreads();
    }
    
    // Write the result to C
    if (row < N && col < M) {
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}