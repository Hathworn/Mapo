#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sgemm_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Calculate global row and column indices
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < M && row < N) { // Ensure within bounds
        float sum = 0.f;
        for (int i = 0; i < K; ++i)
            sum += A[row * K + i] * B[i * M + col]; // Adjusted indexing for B's leading dimension

        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}