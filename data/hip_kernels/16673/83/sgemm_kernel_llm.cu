#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define TILE_DIM 16

__global__ void sgemm_kernel(const float *A, const float *B, float *C, int M, int N, int K, float alpha, float beta) {
    extern __shared__ float shared_mem[];
    float *A_shared = shared_mem;
    float *B_shared = shared_mem + TILE_DIM * TILE_DIM;

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    float element_c = 0.f;

    // Loop over tiles
    for (int tileIdx = 0; tileIdx < (K - 1) / TILE_DIM + 1; ++tileIdx) {
        if (row < M && tileIdx * TILE_DIM + threadIdx.x < K)
            A_shared[threadIdx.y * TILE_DIM + threadIdx.x] = A[row * K + tileIdx * TILE_DIM + threadIdx.x];
        else
            A_shared[threadIdx.y * TILE_DIM + threadIdx.x] = 0.0f;

        if (tileIdx * TILE_DIM + threadIdx.y < K && col < N)
            B_shared[threadIdx.y * TILE_DIM + threadIdx.x] = B[(tileIdx * TILE_DIM + threadIdx.y) * N + col];
        else
            B_shared[threadIdx.y * TILE_DIM + threadIdx.x] = 0.0f;

        __syncthreads();

        // Accumulate results within the shared memory
        for (int e = 0; e < TILE_DIM; e++) {
            element_c += A_shared[threadIdx.y * TILE_DIM + e] * B_shared[e * TILE_DIM + threadIdx.x];
        }
        __syncthreads();
    }

    if (row < M && col < N)
        C[row * N + col] = alpha * element_c + beta * C[row * N + col];
}