#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize by unrolling the loop
__global__ void vecAdd_kernel(float *c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    #pragma unroll 10
    for (int i = 0; i < 500; i++)
        c[idx] = a[idx] + b[idx];
}