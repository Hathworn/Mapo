#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomic_reduction_kernel(float *data_out, float *data_in, int size)
{
    // Calculate global index and check boundary
    int idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx_x < size) {
        atomicAdd(&data_out[0], data_in[idx_x]);
    }
}