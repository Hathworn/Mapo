#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel_A(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if row and col are within bounds
    if (row < N && col < M) {
        float sum = 0.f;

        // Use shared memory for better memory access
        __shared__ float shared_A[32]; // Assuming block size is 32, adjust if different
        __shared__ float shared_B[32]; // Assuming block size is 32, adjust if different

        for (int i = 0; i < K; i += 32) {
            shared_A[threadIdx.y] = A[row * K + i + threadIdx.y];
            shared_B[threadIdx.x] = B[(i + threadIdx.x) * M + col];

            __syncthreads(); // Sync shared memory

            // Use shared memory for computation
            for (int j = 0; j < 32; ++j) {
                sum += shared_A[j] * shared_B[j];
            }

            __syncthreads(); // Ensure complete computation before next iteration
        }

        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}