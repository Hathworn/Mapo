#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    // Calculate global thread row and column 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Use shared memory to improve memory access pattern
    __shared__ float tileA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tileB[BLOCK_SIZE][BLOCK_SIZE];

    float sum = 0.f;
    // Loop over tiles of input matrices
    for (int t = 0; t < (K + BLOCK_SIZE - 1) / BLOCK_SIZE; ++t) {
        if (row < N && t * BLOCK_SIZE + threadIdx.x < K)
            tileA[threadIdx.y][threadIdx.x] = A[row * K + t * BLOCK_SIZE + threadIdx.x];
        else
            tileA[threadIdx.y][threadIdx.x] = 0;

        if (col < M && t * BLOCK_SIZE + threadIdx.y < K)
            tileB[threadIdx.y][threadIdx.x] = B[(t * BLOCK_SIZE + threadIdx.y) * M + col];
        else
            tileB[threadIdx.y][threadIdx.x] = 0;

        __syncthreads();

        for (int i = 0; i < BLOCK_SIZE; ++i)
            sum += tileA[threadIdx.y][i] * tileB[i][threadIdx.x];

        __syncthreads();
    }

    if (row < N && col < M)
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
}