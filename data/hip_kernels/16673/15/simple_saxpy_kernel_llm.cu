#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_saxpy_kernel(float *y, const float* x, const float alpha, const float beta)
{
    // Calculate the global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we are within bounds to avoid out-of-bounds memory access
    if (idx < gridDim.x * blockDim.x) {
        y[idx] = alpha * x[idx] + beta;
    }
}