#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void child_kernel(int *data, int seed)
{
    // Calculate global index only once
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds to avoid out-of-bounds memory access
    if (idx < gridDim.x * blockDim.x) {
        atomicAdd(&data[idx], seed);
    }
}