#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd_kernel(float *c, const float* a, const float* b)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop unrolling for better performance
    #pragma unroll
    for (int i = 0; i < 500; i++)
    {
        c[idx] = a[idx] + b[idx];
    }
}