#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan_v1_kernel(float *d_output, float *d_input, int length)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Initialize element to zero
    float element = 0.f;

    // Use shared memory for partial sums to improve memory access speed
    __shared__ float shared_memory[1024]; // Adjust size based on block size

    // Load data from global to shared memory
    if (idx < length) {
        shared_memory[threadIdx.x] = d_input[idx];
    }
    __syncthreads(); // Synchronize to ensure all loads are complete

    // Perform the scan operation within shared memory
    for (int offset = 0; offset <= threadIdx.x; ++offset) {
        element += shared_memory[threadIdx.x - offset];
    }

    // Write the result back to the output array if within bounds
    if (idx < length) {
        d_output[idx] = element;
    }
}