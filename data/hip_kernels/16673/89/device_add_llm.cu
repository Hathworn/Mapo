#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void device_add(int *a, int *b, int *c) {
    // Calculate global thread index to handle larger data sets
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute c[index] only if within bounds (assuming N is array size)
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}