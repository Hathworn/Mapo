#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void split_kernel(float *d_output, float *d_input, float *d_predicates, float *d_scanned, int length)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if index is within range
    if (idx >= length) return;

    // Use shared memory to minimize global memory access
    __shared__ float shared_input[blockDim.x];
    __shared__ float shared_predicates[blockDim.x];
    __shared__ float shared_scanned[blockDim.x];

    shared_input[threadIdx.x] = d_input[idx];
    shared_predicates[threadIdx.x] = d_predicates[idx];
    shared_scanned[threadIdx.x] = d_scanned[idx];

    __syncthreads();

    if (shared_predicates[threadIdx.x] != 0.f)
    {
        // Calculate address and assign value
        int address = shared_scanned[threadIdx.x] - 1;
        if (address >= 0 && address < length) // Ensure address is within bounds
        {
            d_output[idx] = shared_input[address];
        }
    }
}