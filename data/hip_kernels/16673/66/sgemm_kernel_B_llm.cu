#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemm_kernel_B(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < N && col < M) {  // Ensure row and col are within bounds
        float sum = 0.f;
        for (int i = 0; i < K; ++i) {
            sum += A[row * K + i] * B[i * M + col];  // Corrected indexing for B
        }
        C[row * M + col] = alpha * sum + beta * C[row * M + col];
    }
}