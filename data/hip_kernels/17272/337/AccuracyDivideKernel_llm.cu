#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function to utilize parallelism
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Ensure only one thread performs the division
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}