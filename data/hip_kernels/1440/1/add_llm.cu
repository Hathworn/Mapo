#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory to reduce global memory access latency
    extern __shared__ float shared_x[];
    
    // Load data into shared memory
    if (index < n) {
        shared_x[threadIdx.x] = x[index];
        __syncthreads();  // Synchronize threads within the block

        // Perform computation using shared memory
        for (int i = index; i < n; i += stride) {
            y[i] = shared_x[threadIdx.x] + y[i];
        }
    }
}