#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using device function
__device__ int add_device(int a, int b) {
    return a + b;
}

__global__ void add(int a, int b, int *c)
{
    // Using device function for the addition
    *c = add_device(a, b);
}