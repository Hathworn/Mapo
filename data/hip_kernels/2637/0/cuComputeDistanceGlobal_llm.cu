#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Constants used by the program
#define BLOCK_DIM                      16
#define DEBUG                          0

__global__ void cuComputeDistanceGlobal(float* A, int wA, float* B, int wB, int dim, float* AB) {
    // Shared memory arrays for sub-matrices
    __shared__ float shared_A[BLOCK_DIM][BLOCK_DIM];
    __shared__ float shared_B[BLOCK_DIM][BLOCK_DIM];

    // Thread indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // Calculate block indices
    int block_A = BLOCK_DIM * blockIdx.y;
    int block_B = BLOCK_DIM * blockIdx.x;

    // Initialize squared sum of differences
    float ssd = 0;

    // Loop over all sub-matrices
    for (int a = block_A, b = block_B, end_A = block_A + (dim - 1) * wA; a <= end_A; a += BLOCK_DIM * wA, b += BLOCK_DIM * wB) {
        // Load elements into shared memory
        shared_A[ty][tx] = (a / wA + ty < dim && block_A + tx < wA) ? A[a + wA * ty + tx] : 0;
        shared_B[ty][tx] = (a / wA + ty < dim && block_B + tx < wB) ? B[b + wB * ty + tx] : 0;

        // Synchronize to ensure shared memory is populated
        __syncthreads();

        // Compute differences and accumulate ssd
        if (block_A + ty < wA && block_B + tx < wB) {
            for (int k = 0; k < BLOCK_DIM; ++k) {
                float tmp = shared_A[k][ty] - shared_B[k][tx];
                ssd += tmp * tmp;
            }
        }
        
        // Synchronize before next iteration
        __syncthreads();
    }
    
    // Write results to global memory
    if (block_A + ty < wA && block_B + tx < wB) {
        AB[(block_A + ty) * wB + block_B + tx] = ssd;
    }
}