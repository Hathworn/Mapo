#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Constants used by the program
#define BLOCK_DIM                      16
#define DEBUG                          0

__global__ void cuInsertionSort(float *dist, long *ind, int width, int height, int k) {
    // Variables
    float *p_dist;
    long *p_ind;
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (xIndex < width) {
        // Pointer shift, initialization, and max value
        p_dist = dist + xIndex;
        p_ind = ind + xIndex;
        float max_dist = p_dist[0];
        p_ind[0] = 1;
        
        // Part 1: sort first k elements
        for (int l = 1; l < k; l++) {
            int curr_row = l * width;
            float curr_dist = p_dist[curr_row];
            if (curr_dist < max_dist) {
                int i = l - 1;
                // Use while loop for clarity and efficiency
                while (i >= 0 && p_dist[i * width] > curr_dist) {
                    p_dist[(i+1) * width] = p_dist[i * width];
                    p_ind[(i+1) * width] = p_ind[i * width];
                    i--;
                }
                p_dist[(i+1) * width] = curr_dist;
                p_ind[(i+1) * width] = l + 1;
            } else {
                p_ind[l * width] = l + 1;
            }
            max_dist = p_dist[curr_row];
        }
        
        // Part 2: insert element in the first k lines
        int max_row = (k-1) * width;
        for (int l = k; l < height; l++) {
            float curr_dist = p_dist[l * width];
            if (curr_dist < max_dist) {
                int i = k - 1;
                // Use while loop for clarity and efficiency
                while (i >= 0 && p_dist[i * width] > curr_dist) {
                    p_dist[(i+1) * width] = p_dist[i * width];
                    p_ind[(i+1) * width] = p_ind[i * width];
                    i--;
                }
                p_dist[(i+1) * width] = curr_dist;
                p_ind[(i+1) * width] = l + 1;
                max_dist = p_dist[max_row];
            }
        }
    }
}