#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_DIM                      16
#define DEBUG                          0

__global__ void cuParallelSqrt(float *dist, int width, int k) {
    // Calculate the global index for the current thread
    unsigned int idx = blockIdx.y * width + blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the index is within the valid range
    if (idx < k * width) {
        // Compute the square root of the element at the calculated index
        dist[idx] = sqrt(dist[idx]);
    }
}