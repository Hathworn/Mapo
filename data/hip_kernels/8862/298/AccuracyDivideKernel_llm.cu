#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Check if thread index is valid. Only one thread needed.
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}