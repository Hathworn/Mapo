#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop to cover more elements per thread, improving GPU utilization
    for (int idx = i; idx < numElements; idx += blockDim.x * gridDim.x)
    {
        C[idx] = A[idx] + B[idx];
    }
}