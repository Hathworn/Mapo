#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixTranspose(const float *A_elements, float *B_elements, const int A_width, const int A_height)
{
    // Calculate row and column indices for this thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single condition to check bounds
    if (row < A_width && col < A_height)
    {
        // Perform matrix transpose
        B_elements[row * A_height + col] = A_elements[col * A_width + row];
    }
}