#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VectorAdd(float *VecA, float *VecB, float *VecC, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory for better performance
    extern __shared__ float sharedVecA[];
    extern __shared__ float sharedVecB[];

    // Check if within bounds before accessing
    if (i < size) {
        sharedVecA[threadIdx.x] = VecA[i];
        sharedVecB[threadIdx.x] = VecB[i];
        __syncthreads(); // Ensure all threads have finished writing to shared memory

        // Compute vector addition using shared memory
        VecC[i] = sharedVecA[threadIdx.x] + sharedVecB[threadIdx.x];
    }
}