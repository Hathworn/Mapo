#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    // Use shared memory for faster access
    extern __shared__ float sdata[];

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        // Load data into shared memory
        sdata[threadIdx.x] = A[i] + B[i];

        // Write result from shared memory to global memory
        C[i] = sdata[threadIdx.x];
    }
}