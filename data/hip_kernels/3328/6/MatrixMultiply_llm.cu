#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMultiply(const float* A_elements, const float* B_elements,  float* C_elements, const int X, const int Y, const int Z)
{
    int baseMatrixRow = blockIdx.y * blockDim.y + threadIdx.y;
    int baseMatrixCol = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float As[TILE_SIZE][TILE_SIZE];
    __shared__ float Bs[TILE_SIZE][TILE_SIZE];

    float Cvalue = 0;

    for (int i = 0; i < ((X + TILE_SIZE - 1) / TILE_SIZE); ++i)
    {
        int aRow = baseMatrixRow;
        int aCol = threadIdx.x + i * TILE_SIZE;
        int bRow = threadIdx.y + i * TILE_SIZE;
        int bCol = baseMatrixCol;

        // Load data into shared memory
        if (aRow < Y && aCol < X)
            As[threadIdx.y][threadIdx.x] = A_elements[aRow * X + aCol];
        else
            As[threadIdx.y][threadIdx.x] = 0;

        if (bRow < X && bCol < Z)
            Bs[threadIdx.y][threadIdx.x] = B_elements[bRow * Z + bCol];
        else
            Bs[threadIdx.y][threadIdx.x] = 0;

        __syncthreads();

        // Compute the product using shared memory
        for (int j = 0; j < TILE_SIZE; ++j)
        {
            Cvalue += As[threadIdx.y][j] * Bs[j][threadIdx.x];
        }

        __syncthreads();
    }

    int matrixRow = baseMatrixRow;
    int matrixCol = baseMatrixCol;

    // Save result into global memory
    if (matrixRow < Y && matrixCol < Z)
    {
        C_elements[matrixRow * Z + matrixCol] = Cvalue;
    }
}