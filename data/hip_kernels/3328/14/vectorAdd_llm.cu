#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    // Optimize by using shared memory when beneficial (e.g., for reusing data within a block)
    extern __shared__ float sharedMem[];

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        // Preload elements into shared memory to reduce global memory access (stride could be added if needed)
        sharedMem[threadIdx.x] = A[i] + B[i];
        __syncthreads(); // Ensure all threads have finished loading before updating global memory

        // Write the result back to global memory
        C[i] = sharedMem[threadIdx.x];
    }
}