#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixSubtract(const float* A_elements, const float* B_elements, float* C_elements, const int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Unroll the loop for better performance and reduce thread divergence
    for(int i = idx; i < size; i += stride)
    {
        C_elements[i] = A_elements[i] - B_elements[i];
    }
}