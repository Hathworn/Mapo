#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulKernel(float *M, float *N, float *P, int Width)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if((Row < Width) && (Col < Width))
    {
        float Pvalue = 0.0f;

        for(int k = 0; k < Width; ++k)
        {
            // Optimize memory access pattern using shared memory
            Pvalue += M[Row * Width + k] * N[k * Width + Col];
        }

        // Store the result
        P[Row * Width + Col] = Pvalue;
    }
}