#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void random(unsigned int seed, int* result) {
    // Use shared memory if necessary
    __shared__ hiprandState_t state;

    // Initialize state with thread index and block index for more variability
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state);

    // Generate random number using hiprand and store it at the correct index
    result[idx] = hiprand(&state) % MAX;
}