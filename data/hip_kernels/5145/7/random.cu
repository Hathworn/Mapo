#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void random(unsigned int seed, int* result) {
/* CUDA's random number library uses hiprandState_t to keep track of the seed value
we will store a random state for every thread  */
hiprandState_t state;

/* we have to initialize the state */
hiprand_init(seed, /* the seed controls the sequence of random values that are produced */
0, /* the sequence number is only important with multiple cores */
0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
&state);

/* hiprand works like rand - except that it takes a state as a parameter */
*result = hiprand(&state) % MAX;
}