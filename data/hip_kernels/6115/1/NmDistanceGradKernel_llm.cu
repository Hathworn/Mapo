#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NmDistanceGradKernel(int b, int n, const float *xyz1, int m, const float *xyz2, const float *grad_dist1, const int *idx1, float *grad_xyz1, float *grad_xyz2) {
  // Calculate batch and point indices
  int batch = blockIdx.x;
  int point = threadIdx.x + blockIdx.y * blockDim.x;

  // Ensure thread within bounds
  if (batch < b && point < n) {
    float x1 = xyz1[(batch * n + point) * 3 + 0];
    float y1 = xyz1[(batch * n + point) * 3 + 1];
    float z1 = xyz1[(batch * n + point) * 3 + 2];
    int j2 = idx1[batch * n + point];
    float x2 = xyz2[(batch * m + j2) * 3 + 0];
    float y2 = xyz2[(batch * m + j2) * 3 + 1];
    float z2 = xyz2[(batch * m + j2) * 3 + 2];
    float g = grad_dist1[batch * n + point] * 2;

    // Use atomicAdd for thread safety
    atomicAdd(&(grad_xyz1[(batch * n + point) * 3 + 0]), g * (x1 - x2));
    atomicAdd(&(grad_xyz1[(batch * n + point) * 3 + 1]), g * (y1 - y2));
    atomicAdd(&(grad_xyz1[(batch * n + point) * 3 + 2]), g * (z1 - z2));
    atomicAdd(&(grad_xyz2[(batch * m + j2) * 3 + 0]), -(g * (x1 - x2)));
    atomicAdd(&(grad_xyz2[(batch * m + j2) * 3 + 1]), -(g * (y1 - y2)));
    atomicAdd(&(grad_xyz2[(batch * m + j2) * 3 + 2]), -(g * (z1 - z2)));
  }
}