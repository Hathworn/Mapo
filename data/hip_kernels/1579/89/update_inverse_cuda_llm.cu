#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void update_inverse_cuda(float *Ainv, float *u, int N, int rowstride, int k)
{
    extern __shared__ float shared_mem[];
    float *A_k = shared_mem;
    float *u_shared = &shared_mem[N];
    float *Ainv_u = &shared_mem[2 * N];
    float *Ainv_shared = &shared_mem[3 * N];

    int tid = threadIdx.x;

    // Load data into shared memory
    A_k[tid] = Ainv[k * rowstride + tid];
    u_shared[tid] = u[tid];
    
    // Compute k'th element of Ainv_u using reduction
    float sum = u_shared[tid] * A_k[tid];
    for (int n = blockDim.x >> 1; n > 0; n >>= 1) {
        sum += (tid < n) ? __shfl_down_sync(0xFFFFFFFF, sum, n) : 0.0f;
    }
    if (tid == 0) {
        *Ainv_u = sum;
    }
    __syncthreads();

    float prefact = -1.0f / (1.0f + *Ainv_u);

    // Update each row of Ainv
    for (int row = 0; row < N; row++) {
        Ainv_shared[tid] = Ainv[row * rowstride + tid];
        __syncthreads();

        sum = u_shared[tid] * Ainv_shared[tid];
        for (int n = blockDim.x >> 1; n > 0; n >>= 1) {
            sum += (tid < n) ? __shfl_down_sync(0xFFFFFFFF, sum, n) : 0.0f;
        }
        if (tid == 0) {
            *Ainv_u = sum;
        }
        __syncthreads();

        // Update the value of Ainv for the current row
        Ainv[row * rowstride + tid] = Ainv_shared[tid] + prefact * *Ainv_u * A_k[tid];
        __syncthreads();
    }
}