#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BS 256  // Assuming BS was defined somewhere

__global__ void kernel(float *a, size_t N)
{
    int tid = threadIdx.x;
    int blockIdx = hipBlockIdx_x;
    extern __shared__ float s[];

    // Calculate global thread ID and block count 
    int global_id = blockIdx * blockDim.x + tid;
    float local_sum = 0.0f;

    // Process elements within bounds
    for (int i = global_id; i < N; i += gridDim.x * blockDim.x) {
        s[tid] = a[i];
        __syncthreads();

        // Perform reduction
        for (int skip = BS/2; skip > 0; skip >>= 1) {
            if (tid < skip && tid + skip < N) {
                s[tid] += s[tid + skip];
            }
            __syncthreads();
        }

        if (tid == 0) {
            local_sum += s[0];
        }
    }

    // Perform atomic addition to avoid race condition
    if (tid == 0) {
        atomicAdd(&a[0], local_sum);
    }
}