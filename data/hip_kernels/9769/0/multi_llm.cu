#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multi(float *a, float *b, float *c, int width) {
    // Calculate global thread row and column
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    float result = 0;

    // Check if within matrix bounds
    if (col < width && row < width) {
        // Loop unrolling for better performance
        #pragma unroll
        for (int k = 0; k < width; k++) {
            result += a[row * width + k] * b[k * width + col];
        }
        // Store the computed result
        c[row * width + col] = result;
    }
}