#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc1(float *Ad, float *Bd) {
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    if (tx < 1) {
        float A_temp = Ad[tx]; // Load Ad[tx] into register to reduce memory accesses
        for (int i = 0; i < ITER; i++) {
            A_temp += 1.0f; // Perform arithmetic in register
            __syncthreads(); // Ensure all threads are synchronized if needed
        }
        Ad[tx] = A_temp; // Write the final result back to global memory once
        __syncthreads(); // Ensure Ad[tx] is updated before copying to Bd
        for (int j = 0; j < 256; j++) {
            Bd[tx] = A_temp; // Assign updated A_temp to Bd[tx]
        }
    }
}