#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy(float a, float* x, float* y) {
    // Optimize memory access by computing the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform axpy operation using the computed global index
    y[idx] = a * x[idx];
}