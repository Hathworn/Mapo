#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc2(float *Ad, float *Bd) {
    // Calculate the thread's absolute index
    int tx = threadIdx.x + blockIdx.x * blockDim.x;
    // Ensure threads do not access out-of-bound memory
    if (tx < LEN) {
        // Use a register to cache the value of Ad[tx] during iteration
        float a_val = Ad[tx];
        for (int i = 0; i < ITER; i++) {
            // Increment the cached value
            a_val += 1.0f;
            // Update Bd[tx] directly in the inner loop
            Bd[tx] = a_val;
        }
        // Write back the updated value to Ad after iteration
        Ad[tx] = a_val;
    }
}