#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulSh(float *Md, float *Nd, float *Pd, const int WIDTH)
{
    // Shared memory for tiles
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // Calculate thread coordinates
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;
    float value = 0; // Initialize accumulative sum variable

    for (int m = 0; m < WIDTH / TILE_WIDTH; m++) // Loop over phases
    {
        // Load tiles into shared memory
        Mds[threadIdx.y][threadIdx.x] = Md[row * WIDTH + (m * TILE_WIDTH + threadIdx.x)];
        Nds[threadIdx.y][threadIdx.x] = Nd[(m * TILE_WIDTH + threadIdx.y) * WIDTH + col];
        __syncthreads(); // Synchronize threads

        // Multiply and accumulate
        for (int k = 0; k < TILE_WIDTH; k++)
            value += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        __syncthreads(); // Synchronize threads
    }
    
    // Store the result
    Pd[row * WIDTH + col] = value;
}