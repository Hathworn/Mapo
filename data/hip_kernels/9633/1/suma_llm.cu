#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void suma(int a, int b, int *c) {
    // Use temp variable for result to minimize global memory access
    int sum = a + b;

    // Ensure only one thread modifies *c to avoid data race if multiple blocks are used
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *c = sum;
    }
}