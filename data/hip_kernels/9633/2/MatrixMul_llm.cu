#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMul(float *Md, float *Nd, float *Pd, const int WIDTH) {
    // Calculate thread id
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

    // Initialize accumulation variable
    float Pvalue = 0.0f;

    // Perform the dot product for a single element
    for (int k = 0; k < WIDTH; k++) {
        Pvalue += Md[row * WIDTH + k] * Nd[k * WIDTH + col];
    }

    // Store the result
    Pd[row * WIDTH + col] = Pvalue;
}