#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addOne(int* array, int size) {
    // Use threadIdx.x to increase parallelism, ensuring threads process array within bounds
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        array[index]++;
    }
}