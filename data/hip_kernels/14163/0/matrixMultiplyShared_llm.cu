#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define Tile_size 64

int numARows = 1024;   // number of rows in the matrix A
int numAColumns = 1024;  // number of columns in the matrix A
int numBRows = 1024;   // number of rows in the matrix B
int numBColumns = 1024;  // number of columns in the matrix B
int numCRows = 1024;  // number of rows in the matrix C (you have to set this)
int numCColumns = 1024; // number of columns in the matrix C (you have to set this)

//*************************************************************
__global__ void matrixMultiplyShared(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns)
{
    // Shared memory allocation for sub-matrices
    __shared__ float sA[Tile_size][Tile_size];
    __shared__ float sB[Tile_size][Tile_size];

    // Calculate thread's row and column in global matrix
    int Row = blockDim.y * blockIdx.y + threadIdx.y;
    int Col = blockDim.x * blockIdx.x + threadIdx.x;

    float Cvalue = 0.0f;

    // Loop through tiles of A and B
    for (int k = 0; k < (numAColumns + Tile_size - 1) / Tile_size; ++k)
    {
        // Collaborative loading of A into shared memory
        if (Row < numARows && (k * Tile_size + threadIdx.x) < numAColumns)
        {
            sA[threadIdx.y][threadIdx.x] = A[Row * numAColumns + k * Tile_size + threadIdx.x];
        }
        else
        {
            sA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        // Collaborative loading of B into shared memory
        if (Col < numBColumns && (k * Tile_size + threadIdx.y) < numBRows)
        {
            sB[threadIdx.y][threadIdx.x] = B[(k * Tile_size + threadIdx.y) * numBColumns + Col];
        }
        else
        {
            sB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        // Ensure all threads have loaded their tiles
        __syncthreads();

        // Compute the tile on the shared memory
        for (int j = 0; j < Tile_size; ++j)
        {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }

        // Synchronize to ensure all threads are done computing
        __syncthreads();
    }

    // Write back the result to the global memory
    if (Row < numCRows && Col < numCColumns)
    {
        C[Row * numCColumns + Col] = Cvalue;
    }
}