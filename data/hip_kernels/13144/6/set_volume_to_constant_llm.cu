#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function with coalesced memory access
__global__ void set_volume_to_constant(float * out, float value)
{
    // Calculate 1D thread index for global memory access
    unsigned int thread_index = blockIdx.x * blockDim.x * c_Size.y * c_Size.z +
                                blockIdx.y * blockDim.y * c_Size.z +
                                blockIdx.z * blockDim.z +
                                threadIdx.x * c_Size.y * c_Size.z +
                                threadIdx.y * c_Size.z +
                                threadIdx.z;

    // Ensure thread_index is within array bounds
    if (thread_index < c_Size.x * c_Size.y * c_Size.z) {
        out[thread_index] = value;
    }
}