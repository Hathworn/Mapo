#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gradient_and_subtract_kernel(float *in, float *grad_x, float *grad_y, float *grad_z)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Early exit for out-of-bound indices
    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
        return;
    
    long int id = (k * c_Size.y + j) * c_Size.x + i;

    // Calculate gradients conditionally
    if (i < c_Size.x - 1)
        grad_x[id] -= ((in[id + 1] - in[id]) / c_Spacing.x);
    if (j < c_Size.y - 1)
        grad_y[id] -= ((in[id + c_Size.x] - in[id]) / c_Spacing.y);
    if (k < c_Size.z - 1)
        grad_z[id] -= ((in[id + c_Size.x * c_Size.y] - in[id]) / c_Spacing.z);
}