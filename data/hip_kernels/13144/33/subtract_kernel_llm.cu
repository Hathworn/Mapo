#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtract_kernel(float * in1, float * in2, float * out)
{
    // Calculate 1D flattened thread ID
    unsigned int id = (blockIdx.z * blockDim.z + threadIdx.z) * (c_Size.y * c_Size.x) +
                      (blockIdx.y * blockDim.y + threadIdx.y) * c_Size.x +
                      (blockIdx.x * blockDim.x + threadIdx.x);

    // Early exit if thread ID is out of bounds
    if (threadIdx.x >= c_Size.x || threadIdx.y >= c_Size.y || threadIdx.z >= c_Size.z)
        return;

    // Perform subtraction directly using flattened index
    out[id] = in1[id] - in2[id];
}