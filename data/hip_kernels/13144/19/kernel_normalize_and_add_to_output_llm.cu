#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_normalize_and_add_to_output(float * dev_vol_in, float * dev_vol_out, float * dev_accumulate_weights, float * dev_accumulate_values)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= c_volSize.x || j >= c_volSize.y || k >= c_volSize.z) {
        return; // Early exit for out-of-bounds indices
    }

    long int out_idx = i + (j + k * c_volSize.y) * (c_volSize.x); // Row major indexing
    float eps = 1e-6;
    float acc_weight = dev_accumulate_weights[out_idx]; // Avoid repeated memory access

    if (c_normalize) {
        if (fabsf(acc_weight) > eps) { // Use fabsf to ensure single precision
            dev_vol_out[out_idx] = dev_vol_in[out_idx] + (dev_accumulate_values[out_idx] / acc_weight);
        } else {
            dev_vol_out[out_idx] = dev_vol_in[out_idx]; // Direct assignment without braces
        }
    } else {
        dev_vol_out[out_idx] = dev_vol_in[out_idx] + dev_accumulate_values[out_idx];
    }
}