#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void magnitude_threshold_kernel(float * grad_x, float * grad_y, float * grad_z, float gamma)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Check bounds once using a combined condition
    if (i < c_Size.x && j < c_Size.y && k < c_Size.z) 
    {
        long int id = (k * c_Size.y + j) * c_Size.x + i;
        
        // Compute squared norm to reduce calculations
        float grad_x_val = grad_x[id];
        float grad_y_val = grad_y[id];
        float grad_z_val = grad_z[id];
        float norm_squared = grad_x_val * grad_x_val + grad_y_val * grad_y_val + grad_z_val * grad_z_val;

        // Only calculate sqrt if necessary
        if (norm_squared > gamma * gamma)
        {
            float norm = sqrt(norm_squared);
            float ratio = gamma / norm;
            // Apply thresholding with preloaded values
            grad_x[id] = grad_x_val * ratio;
            grad_y[id] = grad_y_val * ratio;
            grad_z[id] = grad_z_val * ratio;
        }
    }
}