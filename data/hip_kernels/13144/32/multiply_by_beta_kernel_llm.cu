#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiply_by_beta_kernel(float *input, float *output, float beta)
{
    // Calculate global thread ID in 3D space
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;
    
    // Calculate unique linear index for 3D grid
    long int id = (k * c_Size.y + j) * c_Size.x + i;

    // Check for valid index and calculate output if valid
    if (i < c_Size.x && j < c_Size.y && k < c_Size.z) {
        output[id] = input[id] * beta;
    }
}