#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() {
    // Efficiently synchronize all threads within the block
    __syncthreads();
}