#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    // Initialize shared memory with zeros.
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0.0f;

    // Use loop unrolling for each thread.
    if (eidx < gridDim.x * DP_BLOCKSIZE) {
        uint gridSize = gridDim.x * DP_BLOCKSIZE;
        for (; eidx < numElements; eidx += gridSize) {
            shmem[threadIdx.x] += a[eidx] * b[eidx];
        }
    }
    __syncthreads();

    // Optimized reduction using unrolling and avoiding bank conflicts.
    for (int offset = blockDim.x / 2; offset > 32; offset >>= 1) {
        if (threadIdx.x < offset) {
            shmem[threadIdx.x] += shmem[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Handling the last warp separately.
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];

        // Write the result from the first thread of the block to the target.
        if (threadIdx.x == 0) {
            target[blockIdx.x] = mysh[0];
        }
    }
}