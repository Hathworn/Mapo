#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
  // Calculate the index based on thread and block
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  // Perform division only if index is valid to handle parallel execution
  if (index < N) {
    accuracy[index] /= N;
  }
}