#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Compute unique global index for each thread
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    // Use shared memory to enhance memory access patterns if applicable (not shown here due to simplicity)
    // Loop over the array with stride to fully tile tgtWidth * tgtHeight
    for (uint i = idx; i < tgtWidth * tgtHeight; i += stride) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}