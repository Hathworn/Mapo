#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    extern __shared__ float shared_delta[];
    shared_delta[threadIdx.x] = 0;

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            // Accumulate delta into shared memory
            shared_delta[threadIdx.x] += delta[index];
        }
    }

    // Synchronize to ensure all additions are done
    __syncthreads();

    // Use the accumulated value in shared memory
    if (threadIdx.x == 0) {
        float sum_delta = 0;
        for (int t = 0; t < blockDim.x; ++t) {
            sum_delta += shared_delta[t];
        }
        mean_delta[i] = sum_delta * (-1.f / sqrtf(variance[i] + .00001f));
    }
}