#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    // Calculate unique thread index for 1D grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a global stride loop to cover all data
    for (; i < n; i += blockDim.x * gridDim.x)
    {
        // Check mask and apply scaling
        if (mask[i] == mask_num)
        {
            x[i] *= scale;
        }
    }
}