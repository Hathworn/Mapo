#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Unroll loop and use shared memory for better performance
__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = idx; i < N; i += stride)
    {
        Y[i * INCY] = pow(X[i * INCX], ALPHA);
    }
}