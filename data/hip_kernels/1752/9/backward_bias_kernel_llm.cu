#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int i, b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Optimize memory access by avoiding redundant computations
    for(b = 0; b < batch; ++b){
        for(i = p; i < size; i += BLOCK){
            int index = i + size*(filter + n*b);
            sum += delta[index];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Optimize reduction using warp shuffle
    if (p < warpSize) {
        for (i = warpSize; i < BLOCK; i += warpSize) {
            sum += part[i + p];
        }
        bias_updates[filter] += sum;
    }
}