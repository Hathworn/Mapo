#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Use shared memory for reduced global memory access
    extern __shared__ float shared_mem[];
    float *shared_mean = shared_mem;
    float *shared_variance = shared_mean + filters;

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i >= filters) return;

    // Move mean and variance to shared memory
    shared_mean[threadIdx.x] = mean[i];
    shared_variance[threadIdx.x] = variance[i];
    __syncthreads();

    float sum = 0.0f; // Local accumulator for variance_delta

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            // Calculate sum
            sum += delta[index] * (x[index] - shared_mean[threadIdx.x]);
        }
    }

    // Final computation using shared variance
    variance_delta[i] = sum * (-0.5f * powf(shared_variance[threadIdx.x] + 0.000001f, -1.5f));
}