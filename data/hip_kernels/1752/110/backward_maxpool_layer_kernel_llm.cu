#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    // Calculate commonly used indices
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n) return;

    int w_offset = -pad;
    int h_offset = -pad;
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;
    int area = (size - 1) / stride;

    // Compute indices for current thread
    int index = idx;
    int j = index % in_w;
    index /= in_w;
    int i = index % in_h;
    index /= in_h;
    int k = index % in_c;
    index /= in_c;
    int b = index;

    float d = 0.0f;
    for (int l = -area; l <= area; ++l) {
        for (int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            int out_index = out_w + w * (out_h + h * (k + c * b));

            // Check validity and accumulate delta if valid
            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h) {
                if (indexes[out_index] == idx) {
                    d += delta[out_index];
                }
            }
        }
    }
    prev_delta[idx] += d; // Accumulate result into prev_delta
}