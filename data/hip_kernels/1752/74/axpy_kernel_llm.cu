#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX,  float *Y, int OFFY, int INCY)
{
    // Optimize access by precalculating offsets
    int idxX = OFFX + threadIdx.x + blockIdx.x * blockDim.x;
    int idxY = OFFY + threadIdx.x + blockIdx.x * blockDim.x;

    // Check bounds inside the loop
    if(idxX * INCX < N && idxY * INCY < N) {
        Y[idxY * INCY] += ALPHA * X[idxX * INCX]; // Perform computation
    }
}