#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * (gridDim.x * blockDim.x);
    
    // Use a stride loop to handle more elements per thread
    for(int idx = i; idx < N; idx += gridDim.x * blockDim.x * gridDim.y)
    {
        Y[idx*INCY + OFFY] = X[idx*INCX + OFFX];
    }
}