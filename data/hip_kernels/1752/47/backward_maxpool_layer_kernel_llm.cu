#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    // Calculate output dimensions
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;
    int area = (size - 1) / stride;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % in_w;
    int i = (id / in_w) % in_h;
    int k = (id / in_w / in_h) % in_c;
    int b = id / in_w / in_h / in_c;

    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0;

    // Use global memory read coalescing to optimize memory access
    for (int l = -area; l <= area; ++l) {
        for (int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;

            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h) {
                int out_index = out_w + w * (out_h + h * (k + c * b));
                if (indexes[out_index] == id) {
                    d += delta[out_index];
                }
            }
        }
    }

    prev_delta[id] += d;
}
```
