#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    // Calculate global index using blockIdx, blockDim, and threadIdx
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop by using a stride
    for (; i < n; i += blockDim.x * gridDim.x) {
        // Apply mask condition and set value
        if (mask[i] == mask_num) {
            x[i] = val;
        }
    }
}