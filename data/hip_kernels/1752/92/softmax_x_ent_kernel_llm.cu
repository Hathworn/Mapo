#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate unique global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop to handle all elements (more efficient for large n)
    for (int i = idx; i < n; i += blockDim.x * gridDim.x) {
        float t = truth[i];
        float p = pred[i];
        // Optimize conditional assignment and operation
        error[i] = t ? -log(p) : 0.0f;
        delta[i] = t - p;
    }
}