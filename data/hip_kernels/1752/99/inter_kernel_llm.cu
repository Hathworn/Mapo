#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate global index of the thread within the grid
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    // Total number of elements in the output array
    int total_elements = (NX + NY) * B;

    // Loop over multiple elements if the grid size is small
    for(int index = i; index < total_elements; index += blockDim.x * gridDim.x) {
        int b = index / (NX + NY); // Determine block number in batch
        int j = index % (NX + NY); // Determine local index within block

        // Copy from X array if within NX, else from Y array
        OUT[index] = (j < NX) ? X[b * NX + j] : Y[b * NY + j - NX];
    }
}