#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Using blockDim.x and blockDim.y to improve memory access patterns
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = gridDim.x * blockDim.x; // Calculation once to reduce arithmetic

    // Loop to maximize utilization and prevent branching
    for (int i = idx; i < N; i += gridSize) {
        X[i * INCX] = ALPHA;
    }
}