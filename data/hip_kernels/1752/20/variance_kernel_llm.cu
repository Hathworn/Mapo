#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Optimize by precomputing constant in registers
    float scale = 1.0f / (batch * spatial - 1);
    int idx = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (idx >= filters) return;

    float var_sum = 0.0f; // Use local variable to accumulate variance

    for (int j = 0; j < batch; ++j) {
        // Unroll loop to improve performance
        for (int k = 0; k < spatial; k += 4) {
            int base_index = j * filters * spatial + idx * spatial + k;

            for (int offset = 0; offset < 4 && (k + offset) < spatial; ++offset) {
                int index = base_index + offset;
                float diff = x[index] - mean[idx];
                var_sum += diff * diff;
            }
        }
    }
    variance[idx] = var_sum * scale; // Store result
}