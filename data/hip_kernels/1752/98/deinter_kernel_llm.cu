#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = gridDim.x * blockDim.x; // Total number of threads
    while (i < (NX + NY) * B) {
        int b = i / (NX + NY);
        int j = i % (NX + NY);

        // Update value in X if j is within NX
        if (j < NX) {
            if (X) {
                X[b * NX + j] += OUT[i];
            }
        } else { // Update value in Y if j is >= NX
            if (Y) {
                Y[b * NY + j - NX] += OUT[i];
            }
        }
        i += gridSize; // Move to next relevant index
    }
}