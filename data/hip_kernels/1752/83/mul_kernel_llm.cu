#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global thread index for 1D grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use this condition to ensure access within bounds
    if (i < N) {
        // Direct access and operation on Y using calculated index
        Y[i * INCY] *= X[i * INCX];
    }
}