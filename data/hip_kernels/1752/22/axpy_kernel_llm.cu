#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation

    // Looping with stride to ensure each thread processes multiple elements if necessary
    for (; i < N; i += blockDim.x * gridDim.x) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}