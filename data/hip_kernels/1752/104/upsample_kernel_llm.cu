#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified to 1D grid
    if(i >= N) return;

    int out_index = i; // Keep out_index to set the output
    int s = stride * stride;
    int offset = out_index % s; 

    int out_w = (out_index / s) % w;
    int out_h = (out_index / (s * w)) % h;
    int out_c = (out_index / (s * w * h)) % c;
    int b = out_index / (s * w * h * c);
    
    int in_w = out_w;
    int in_h = out_h;
    int in_c = out_c;

    int in_index = b * w * h * c + in_c * w * h + in_h * w + in_w; // Compute input index

    if(forward) 
        out[out_index] += scale * x[in_index];
    else 
        atomicAdd(x + in_index, scale * out[out_index]);
}