#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify index calculation for 1D grid
    if(i < n) {
        float s_val = s[i];                         // Pre-load s[i] into register
        float b_val = b ? b[i] : 0;                 // Pre-load b[i] into register if b is not NULL
        c[i] = s_val * a[i] + (1 - s_val) * b_val;  // Avoid repeated calculation of (1-s[i])
    }
}