#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate linear index
    int stride = blockDim.x * gridDim.x; // Determine grid stride
    for (; i < N; i += stride) { // Use grid-stride loop
        Y[i * INCY] *= X[i * INCX]; // Perform multiplication
    }
}