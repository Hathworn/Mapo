#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    // Calculate the flat index for the current thread
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    
    int b = index / spatial;
    int i = index % spatial;
    float sum = 0.0f;
    
    // Reduce and cache sum of squares using registers
    for (int f = 0; f < filters; ++f) {
        int idx = b * filters * spatial + f * spatial + i;
        float val = x[idx];
        sum += val * val;
    }
    
    // Normalize sum and guard against division by zero
    sum = sqrtf(sum);
    float invSum = (sum == 0.0f) ? 1.0f : 1.0f / sum;
    
    // Normalize x and compute dx using precomputed invSum
    for (int f = 0; f < filters; ++f) {
        int idx = b * filters * spatial + f * spatial + i;
        x[idx] *= invSum;
        dx[idx] = (1.0f - x[idx]) * invSum;
    }
}