#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate unique global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit if index is out of range
    if (i >= N) return;

    // Efficient computing of indices using modulo and division operations
    int in_s = i % spatial;
    int in_c = (i / spatial) % layers;
    int b = i / (spatial * layers); // Calculate batch index directly

    // Calculate in and out indices
    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;

    // Conditional assignment
    if (forward) 
        out[i2] = x[i1];
    else 
        out[i1] = x[i2];
}