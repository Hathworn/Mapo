#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index for the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;  // Calculate the stride
    
    // Use stride to allow each thread to compute multiple elements
    for (int index = i; index < N; index += stride) {
        X[index * INCX] *= ALPHA;  // Scale the vector element
    }
}