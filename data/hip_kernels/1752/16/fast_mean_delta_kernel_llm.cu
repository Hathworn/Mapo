#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0.0f;

    int filter = blockIdx.x;

    // Coalesced memory access
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += delta[index];
        }
    }

    // Use reduction to optimize the summation
    __syncthreads();
    for (int stride = threads / 2; stride > 0; stride >>= 1) {
        if (id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    // Write the result to global memory
    if (id == 0) {
        mean_delta[filter] = local[0] * (-1.0f / sqrtf(variance[filter] + 0.000001f));
    }
}