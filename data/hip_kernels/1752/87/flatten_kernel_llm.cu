#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= N) return;
    
    int b = i / (layers * spatial); // Calculate the batch index
    int index = i % (layers * spatial); // Compute remaining index within batch
    int in_c = index / spatial; // Calculate channel index
    int in_s = index % spatial; // Calculate spatial index

    int i1 = b * layers * spatial + in_c * spatial + in_s; // Input index calculation
    int i2 = b * layers * spatial + in_s * layers + in_c; // Output index calculation

    // Efficiently map input to output based on forward flag
    if (forward)
        out[i2] = x[i1];
    else
        out[i1] = x[i2];
}