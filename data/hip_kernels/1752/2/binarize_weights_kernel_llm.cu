#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    
    // Use register variable for mean to reduce global memory access
    float mean = 0.0f;
    
    // Optimize loop by using float2 vectorized loads
    for (int i = 0; i < size; i += 2)
    {
        float2 weight_pair = reinterpret_cast<float2*>(weights + f * size + i)[0];
        mean += fabs(weight_pair.x) + fabs(weight_pair.y);
    }

    // Compute mean value
    mean = mean / size;
    
    // Optimize loop by using float2 vectorized stores
    for (int i = 0; i < size; i += 2)
    {
        float2 weight_pair = reinterpret_cast<float2*>(weights + f * size + i)[0];
        float2 binary_pair;
        binary_pair.x = (weight_pair.x > 0) ? mean : -mean;
        binary_pair.y = (weight_pair.y > 0) ? mean : -mean;
        reinterpret_cast<float2*>(binary + f * size + i)[0] = binary_pair;
    }
}