#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index >= N) return;

    // Calculate input positions
    int b = index / (w * h * c);
    int in_index = index % (w * h * c);
    int in_c = (in_index / (w * h)) % c;
    int in_h = (in_index / w) % h;
    int in_w = in_index % w;

    // Calculate output component dimensions
    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;

    // Calculate output index
    int out_index = b * (out_c * h * stride * w * stride) 
                    + c2 * (h * stride * w * stride) 
                    + h2 * (w * stride) 
                    + w2;

    // Perform reorganization
    if (forward) {
        out[out_index] = x[index];
    } else {
        out[index] = x[out_index];
    }
}