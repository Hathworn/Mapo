#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Calculate unique thread index
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;

    // Early exit if f2 <= f1
    if (f2 <= f1) return;

    float sum = 0.0f;
    float norm1 = 0.0f;
    float norm2 = 0.0f;

    // Loop through batch and size to calculate sums and norms
    for(int b = 0; b < batch; ++b){
        for(int i = 0; i < size; ++i){
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            float val1 = output[i1];
            float val2 = output[i2];
            sum += val1 * val2;
            norm1 += val1 * val1;
            norm2 += val2 * val2;
        }
    }
    
    // Calculate norms outside of loops
    norm1 = sqrtf(norm1);
    norm2 = sqrtf(norm2);
    float norm = norm1 * norm2;

    // Compute updates for delta using precalculated norm and sum
    float scaled_sum = -scale * sum / norm;
    for(int b = 0; b < batch; ++b){
        for(int i = 0; i < size; ++i){
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            delta[i1] += scaled_sum * output[i2];
            delta[i2] += scaled_sum * output[i1];
        }
    }
}