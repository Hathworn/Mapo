#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    // Calculate correct index instead of manually computing offset
    int index = (blockIdx.z * n + blockIdx.y) * size + blockIdx.x * blockDim.x + threadIdx.x;

    // Use a conditional check to ensure we don't exceed the limits
    if(index < (n * size)) {
        output[index] += biases[blockIdx.y];
    }
}