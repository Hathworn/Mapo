#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    // Calculate global index
    int s = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if index out of bounds
    if (s >= size) return;

    // Use shared memory for input tile to reduce global memory accesses
    __shared__ float shared_input[1024];  // Maximum shared memory usage depends on the GPU, adjust size as needed

    float mean = 0;

    // Loop over 'n' using block-stride loop
    for (int i = 0; i < n; ++i) {
        // Load input into shared memory
        shared_input[threadIdx.x] = abs(input[i * size + s]);
        __syncthreads();

        // Accumulate mean value
        mean += shared_input[threadIdx.x];

        // Synchronize to ensure shared memory is consistent before next iteration
        __syncthreads();
    }

    // Calculate mean
    mean /= n;

    // Reiterate for binarization
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}