#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Cache grid dimensions as constant expressions
    const int gridWidth = gridDim.x;
    const int blockWidth = blockDim.x;
    
    // Calculate global index
    int i = (blockIdx.x + blockIdx.y * gridWidth) * blockWidth + threadIdx.x;

    // Perform operation only if index is within bounds
    if (i < N) {
        float value = X[i * INCX];
        // Utilize clamps to optimize the constraint calculation
        X[i * INCX] = clamp(value, -ALPHA, ALPHA);
    }
}