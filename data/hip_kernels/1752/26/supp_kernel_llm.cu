#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index once for reuse
    int index = i * INCX;
    
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if(i < N) {
        float val = X[index]; // Load value once
        if((val * val) < (ALPHA * ALPHA)) {
            X[index] = 0; // Set to zero directly
        }
    }
}