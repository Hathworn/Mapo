#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int i, b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f;

    // Loop unrolling for performance
    for(b = 0; b < batch; ++b) {
        for(i = 0; i < size; i += BLOCK * 4) {
            int index0 = p + i + size * (filter + n * b);
            int index1 = p + i + BLOCK + size * (filter + n * b);
            int index2 = p + i + 2 * BLOCK + size * (filter + n * b);
            int index3 = p + i + 3 * BLOCK + size * (filter + n * b);

            if (p + i < size) sum += delta[index0] * x_norm[index0];
            if (p + i + BLOCK < size) sum += delta[index1] * x_norm[index1];
            if (p + i + 2 * BLOCK < size) sum += delta[index2] * x_norm[index2];
            if (p + i + 3 * BLOCK < size) sum += delta[index3] * x_norm[index3];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Using parallel reduction for better performance
    if (p < 32) {
        part[p] += part[p + 32];
        part[p] += part[p + 16];
        part[p] += part[p + 8];
        part[p] += part[p + 4];
        part[p] += part[p + 2];
        part[p] += part[p + 1];
    }

    // Single thread update for scale_updates
    if (p == 0) {
        scale_updates[filter] += part[0];
    }
}