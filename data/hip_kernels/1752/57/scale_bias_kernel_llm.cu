#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    // Calculate the global index
    int idx = blockIdx.z * n * size + blockIdx.y * size + blockDim.x * blockIdx.x + threadIdx.x;
    
    // Perform operation only if within bounds
    if(idx < n * size) {
        output[idx] *= biases[blockIdx.y];
    }
}