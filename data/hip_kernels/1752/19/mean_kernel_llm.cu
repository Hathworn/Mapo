#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.0f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Exit early if thread is outside the valid range
    if (i >= filters) return;

    float sum = 0.0f; // Local accumulator for mean calculation
    int base_index = i * spatial; // Precompute base index for efficiency
    for (int j = 0; j < batch; ++j) {
        int batch_base_index = j * filters * spatial + base_index; // Avoid repeated calculation
        for (int k = 0; k < spatial; ++k) {
            sum += x[batch_base_index + k]; // Efficient accumulation
        }
    }
    mean[i] = sum * scale; // Store the result with scaling
}