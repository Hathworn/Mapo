#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = gridDim.x * blockDim.x;

    // Use loop to parallelize for more work if thread count is less than size
    for (int idx = id; idx < size; idx += total_threads) {
        int i = idx % minw;
        int idx_temp = idx / minw;
        int j = idx_temp % minh;
        idx_temp /= minh;
        int k = idx_temp % minc;
        idx_temp /= minc;
        int b = idx_temp % batch;

        // Calculate indexes for input and output arrays
        int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
        int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));

        // Perform addition
        out[out_index] += add[add_index];
    }
}