#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Use shared memory for improved memory access
    // Allocate shared memory for block calculations
    extern __shared__ float shared_mem[];

    // Calculate the index and load data into shared memory
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float diff = 0.0f, abs_val = 0.0f;

    if (i < n) {
        diff = truth[i] - pred[i];
        abs_val = fabsf(diff);

        // Utilize shared memory for intermediate calculations
        shared_mem[threadIdx.x] = abs_val;

        __syncthreads();  // Synchronize within block to ensure shared memory is updated

        if(shared_mem[threadIdx.x] < 1) {
            error[i] = diff * diff;
            delta[i] = diff;
        } else {
            error[i] = 2 * shared_mem[threadIdx.x] - 1;
            delta[i] = (diff > 0) ? 1 : -1;
        }
    }
}