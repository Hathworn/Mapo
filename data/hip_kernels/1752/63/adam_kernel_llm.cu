#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    // Calculate the global linear index
    int index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Check if the current index is within bounds
    if (index < N) {
        // Precompute constants for efficiency
        float B1_t = powf(B1, t);
        float B2_t = powf(B2, t);
        
        // Calculate mhat and vhat using precomputed terms
        float mhat = m[index] / (1.f - B1_t);
        float vhat = v[index] / (1.f - B2_t);
        
        // Update the x array using computed terms
        x[index] += rate * mhat / (sqrtf(vhat) + eps);
    }
}