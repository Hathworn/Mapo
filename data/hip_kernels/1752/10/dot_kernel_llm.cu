#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Calculate global index for unique thread identification
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;

    // Exit if f2 is not greater than f1
    if (f2 <= f1) return;

    // Initialize variables for sum and norms
    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;

    // Iterate over the batch and size to compute sum and norms
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            float val1 = output[i1];
            float val2 = output[i2];

            // Accumulate sum and norms
            sum += val1 * val2;
            norm1 += val1 * val1;
            norm2 += val2 * val2;
        }
    }

    // Calculate the norms and adjust the sum
    norm1 = rsqrtf(norm1);  // Use fast rsqrt approximation
    norm2 = rsqrtf(norm2);  // Use fast rsqrt approximation
    float norm = norm1 * norm2;
    sum *= norm;

    // Update delta values with the computed scale, sum, and output adjustments
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            float scaledSum = scale * sum;
            atomicAdd(&delta[i1], -scaledSum * output[i2]);
            atomicAdd(&delta[i2], -scaledSum * output[i1]);
        }
    }
}