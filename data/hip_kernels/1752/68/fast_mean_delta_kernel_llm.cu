#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    // Loop unrolling for spatial dimension
    for (j = 0; j < batch; ++j) {
        for (i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += delta[index];
        }
    }

    __syncthreads();

    // Use parallel reduction to sum local array
    if (id < 32) {
        for (i = 32; i < threads; i += 32) {
            local[id] += local[id + i];
        }
    }

    if (id == 0) {
        mean_delta[filter] = 0;
        for (i = 0; i < 32; ++i) {
            mean_delta[filter] += local[i];
        }
        mean_delta[filter] *= (-1.f / sqrtf(variance[filter] + .00001f));
    }
}