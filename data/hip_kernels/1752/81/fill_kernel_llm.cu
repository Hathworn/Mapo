#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 
              (blockIdx.y * gridDim.x) * (blockDim.x * gridDim.y);

    // Use a stride loop for coalesced memory access
    for (int i = idx; i < N; i += blockDim.x * gridDim.x * gridDim.y)
    {
        X[i * INCX] = ALPHA;
    }
}