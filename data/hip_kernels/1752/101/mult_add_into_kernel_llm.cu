#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds and perform computation
    if(i < n){
        c[i] += a[i] * b[i];
    }
}