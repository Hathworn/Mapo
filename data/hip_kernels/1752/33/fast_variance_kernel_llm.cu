#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[BLOCK];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0.0f;
    
    // Optimize loop unrolling and avoid unnecessary computations
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            float diff = x[index] - mean[filter];
            local[id] += diff * diff;
        }
    }

    // Use parallel reduction to sum shared memory
    __syncthreads();
    for (int stride = threads / 2; stride > 0; stride >>= 1) {
        if (id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    // Final compute on thread 0
    if (id == 0) {
        variance[filter] = local[0] / (spatial * batch - 1);
    }
}