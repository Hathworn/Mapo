#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n) {
        // Load data into registers
        float truth_val = truth[i];
        float pred_val = pred[i];

        // Compute difference
        float diff = truth_val - pred_val;
        
        // Write results back to global memory
        error[i] = diff * diff;
        delta[i] = diff;
    }
}