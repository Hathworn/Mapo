#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Use more efficient thread index calculation
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i < n) {
        float diff = truth[i] - pred[i];
        error[i] = fabsf(diff); // Use faster intrinsic function for absolute value
        delta[i] = copysignf(1.0f, diff); // Use copysign function instead of branch
    }
}