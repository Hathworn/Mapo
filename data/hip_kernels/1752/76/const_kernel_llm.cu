#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x; // Calculate global thread index
    int stride = gridDim.x * gridDim.y * blockDim.x; // Define stride for threads
    for(int i = idx; i < N; i += stride) // Use stride to cover all elements
    {
        X[i * INCX] = ALPHA; // Perform the operation
    }
}