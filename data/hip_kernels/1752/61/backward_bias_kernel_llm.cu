#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Loop unrolling for better performance
    for(int b = 0; b < batch; ++b){
        for(int i = 0; i < size; i += BLOCK * 2){
            int index1 = p + i + size*(filter + n*b);
            int index2 = p + i + BLOCK + size*(filter + n*b);
            sum += (p+i < size) ? delta[index1] : 0;
            sum += (p+i + BLOCK < size) ? delta[index2] : 0;
        }
    }
    
    part[p] = sum;
    __syncthreads();
    
    // Reduction step optimization
    if (p < BLOCK / 2) {
        part[p] += part[p + BLOCK / 2];
    }
    __syncthreads();
    
    if (p == 0) {
        for(int i = 0; i < BLOCK / 2; ++i) bias_updates[filter] += part[i];
    }
}