#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    // Calculate global thread index efficiently
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        // Precompute constants outside of the indexing
        float one_minus_B2t = 1.0f - __powf(B2, t);
        float one_minus_B1t = 1.0f - __powf(B1, t);

        // Update x[index] using precomputed constants for better performance
        float denom = sqrtf(v[index]) + eps;
        float update = (rate * sqrtf(one_minus_B2t) / one_minus_B1t) * (m[index] / denom);

        x[index] -= update;
    }
}