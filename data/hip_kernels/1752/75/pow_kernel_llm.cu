#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for grid-stride loop
    for(; i < N; i += stride) { // Use grid-stride loop for better occupancy
        Y[i * INCY] = powf(X[i * INCX], ALPHA); // Use powf for single-precision floating-point
    }
}