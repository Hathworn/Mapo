#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    float sum = 0.0f;
    float largest = -INFINITY;
    
    // Use shared memory for the largest element
    extern __shared__ float shared[];
    float *shared_largest = shared;
    int tid = threadIdx.x;

    // Parallel reduction to find the largest element
    for (int i = tid; i < n; i += blockDim.x) {
        float val = input[i*stride];
        atomicMax(shared_largest, val);
    }
    __syncthreads();
    largest = *shared_largest;

    // Calculate exponentials and compute sum
    for (int i = tid; i < n; i += blockDim.x) {
        float e = expf((input[i*stride] - largest) / temp);
        atomicAdd(&sum, e);
        output[i*stride] = e;
    }
    __syncthreads();

    // Normalize outputs
    for (int i = tid; i < n; i += blockDim.x) {
        output[i*stride] /= sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= batch * groups) return;

    int b = id / groups;
    int g = id % groups;

    // Launch softmax_device with dynamically allocated shared memory
    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}