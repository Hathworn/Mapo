#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[BLOCK];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;
    int total_spatial = batch * spatial * filters;

    // Using stride loop to reduce iteration count
    for (int index = filter * spatial + id; index < total_spatial; index += threads * spatial) {
        int j = index / (spatial * filters);
        int i = index % spatial;
        local[id] += (i < spatial) ? delta[index] * (x[index] - mean[filter]) : 0;
    }

    __syncthreads();

    // Using reduction to minimize warp divergence
    for (int offset = threads / 2; offset > 0; offset >>= 1) {
        if (id < offset) {
            local[id] += local[id + offset];
        }
        __syncthreads();
    }

    // Only first thread writes the result
    if (id == 0) {
        variance_delta[filter] = local[0] * -.5f * powf(variance[filter] + .00001f, -1.5f);
    }
}