#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (i < n) {
        float t = truth[i];
        float p = pred[i];
        // Precompute log values to improve readability and performance
        float log_p = logf(p + 1e-7f);
        float log_1mp = logf(1 - p + 1e-7f);
        
        error[i] = -t * log_p - (1.0f - t) * log_1mp;
        delta[i] = t - p;
    }
}