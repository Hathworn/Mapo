#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial) {
    int index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x; // Optimize indexing calculation
    if (index < N) {
        int f = (index / spatial) % filters; // Calculate filter index
        float var = variance[f] + 1e-5f; // Pre-compute variance term
        x[index] = (x[index] - mean[f]) / sqrtf(var); // Normalize
    }
}