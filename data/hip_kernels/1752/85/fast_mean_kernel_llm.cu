#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Unrolling the loop for better performance
    for(int j = 0; j < batch; ++j){
        for(int i = id; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i;
            local[id] += x[index];
        }
    }

    __syncthreads();

    // Using parallel reduction to calculate the sum
    if (id == 0) {
        mean[filter] = 0;
        for (int i = 0; i < threads; ++i) {
            mean[filter] += local[i];
        }
        mean[filter] /= spatial * batch;
    }
}