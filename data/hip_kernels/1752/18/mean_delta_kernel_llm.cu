#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float mean_sum = 0.0f; // Initialize accumulator
    int total_elements = batch * spatial;

    // Unroll loop to efficiently compute mean_sum for batches and spatial elements
    for (int j = 0; j < total_elements; ++j) {
        int index = j * filters + i;
        mean_sum += delta[index];
    }

    mean_delta[i] = mean_sum * (-1.0f / sqrtf(variance[i] + 0.000001f)); // Pre-calculate factor outside the loop
}