#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll grid manually for better utilization
    int stride = gridDim.x * blockDim.x;
    for (; i < n; i += stride)
    {
        // Compute weighted sum
        float b_val = b ? b[i] : 0.0f; // Conditional retrieval
        c[i] = s[i] * a[i] + (1.0f - s[i]) * b_val;
    }
}