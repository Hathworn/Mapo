#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    // Find the largest value for numerical stability
    for(i = 0; i < n; ++i){
        float val = input[i*stride];
        largest = fmaxf(val, largest); // Use fmaxf for better performance
    }
    for(i = 0; i < n; ++i){
        float e = expf(input[i*stride]/temp - largest/temp);
        sum += e;
        output[i*stride] = e;
    }
    for(i = 0; i < n; ++i){
        output[i*stride] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    if (id >= spatial*batch*groups) return;

    int s = id % spatial;
    id = id / spatial;
    int g = id % groups;
    int b = id / groups;

    int goff = group_offset[g]*spatial;
    int boff = b*stride;

    // Process softmax computation for this specific slice
    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}