#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread ID in a more efficient way
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the operation only happens for threads within bounds
    if (i < N) 
    {
        X[i * INCX] *= ALPHA;
    }
}