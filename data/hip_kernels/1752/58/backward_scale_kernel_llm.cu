#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Use unrolled loop to improve memory access
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; i += BLOCK) {
            int index = p + i + size * (filter + n * b);
            if (p + i < size) sum += delta[index] * x_norm[index];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Parallel reduction instead of sequential reduction
    for (int stride = BLOCK / 2; stride > 0; stride /= 2) {
        if (p < stride) {
            part[p] += part[p + stride];
        }
        __syncthreads();
    }

    if (p == 0) {
        atomicAdd(&scale_updates[filter], part[0]);
    }
}