#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    // Compute global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the index is within bounds
    if (id >= size) return;
    
    // Unroll loop to directly compute indices
    int b = id / (minw * minh * minc);
    int k = (id / (minw * minh)) % minc;
    int j = (id / minw) % minh;
    int i = id % minw;
    
    // Compute output and add indices efficiently
    int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));
    
    // Accumulate results
    out[out_index] = s1 * out[out_index] + s2 * add[add_index];
}