#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation
    int stride = gridDim.x * blockDim.x; // Calculate stride for grid-stride loop

    for (; i < n; i += stride) { // Use grid-stride loop for better memory coalescing
        // Use conditional operator for concise branching
        if (da) da[i] += dc[i] * s[i]; 
        db[i] += dc[i] * (1-s[i]);
        ds[i] += dc[i] * (a[i] - b[i]); // Simplify operation
    }
}