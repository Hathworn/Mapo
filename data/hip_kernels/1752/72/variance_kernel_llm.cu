#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Pre-calculate constants outside loops for reduced computation.
    float scale = 1.f / (batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for variance accumulation.
    __shared__ float shared_variance[1024];
    if (threadIdx.x < filters) shared_variance[threadIdx.x] = 0.0f; 
    __syncthreads();

    if (i < filters) {
        for (int j = 0; j < batch; ++j) {
            for (int k = 0; k < spatial; ++k) {
                int index = j * filters * spatial + i * spatial + k;
                shared_variance[threadIdx.x] += powf((x[index] - mean[i]), 2);
            }
        }
    }
    __syncthreads();

    // Write accumulated variance back to global memory with scale adjustment.
    if (i < filters) {
        variance[i] = shared_variance[threadIdx.x] * scale;
    }
}