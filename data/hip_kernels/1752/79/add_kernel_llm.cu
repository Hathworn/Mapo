#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D grid and block for simplicity
    int stride = gridDim.x * blockDim.x;          // Calculate stride for loop-based processing
    for (; i < N; i += stride)                    // Loop through elements with stride
    {
        X[i * INCX] += ALPHA;
    }
}