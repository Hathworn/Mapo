#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Calculate global thread index
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    
    // Return if the index is out of range
    if (i >= groups) return;
    
    // Use local variable to accumulate results to reduce memory access
    float local_sum = 0.0f;
    
    // Perform accumulation
    for(int k = 0; k < n; ++k){
        local_sum += x[k * groups + i];
    }
    
    // Write back to global memory
    sum[i] = local_sum;
}