#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop over elements with stride to utilize all threads
    for (int i = index; i < N; i += stride) {
        int f = (i / spatial) % filters;
        x[i] = (x[i] - mean[f]) / (sqrt(variance[f]) + 0.000001f);
    }
}