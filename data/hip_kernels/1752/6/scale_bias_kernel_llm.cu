#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Check if offset is within bounds and perform scaling
    if(offset < size)
    {
        int idx = (batch * n + filter) * size + offset;
        output[idx] *= biases[filter];
    }
}