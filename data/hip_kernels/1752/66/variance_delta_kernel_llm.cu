#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Precompute inverse power of variance for efficiency
    float inv_var = -.5f * powf(variance[i] + .00001f, -1.5f);
    float sum = 0; // Accumulate contributions in a local variable
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += delta[index] * (x[index] - mean[i]); // Use local variable for accumulation
        }
    }
    variance_delta[i] = sum * inv_var; // Use precomputed inverse power
}