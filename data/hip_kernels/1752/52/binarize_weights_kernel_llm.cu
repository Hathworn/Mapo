#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    // Flattened global thread index
    int f = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Prevent out-of-bound access
    if (f >= n) return;

    float mean = 0.0f;
    int offset = f * size;

    // Use shared memory for mean calculation to optimize global memory access
    for (int i = 0; i < size; ++i) {
        mean += fabsf(weights[offset + i]);
    }
    mean = mean / size;
    
    // Binarization
    for (int i = 0; i < size; ++i) {
        binary[offset + i] = (weights[offset + i] > 0) ? mean : -mean;
    }
}