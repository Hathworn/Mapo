#include "hip/hip_runtime.h"
```cuda
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    // Calculate unique index for each thread
    int s = blockIdx.x * blockDim.x + threadIdx.x;
    if (s >= size) return;

    float mean = 0.0f;

    // Use shared memory for intermediate mean calculation
    extern __shared__ float shared_mem[];
    for (int i = 0; i < n; ++i) {
        shared_mem[threadIdx.x] = fabsf(input[i * size + s]);
        __syncthreads();

        // Accumulate in shared memory
        if (threadIdx.x == 0) {
            float block_sum = 0.0f;
            for (int j = 0; j < blockDim.x; ++j) {
                block_sum += shared_mem[j];
            }
            mean += block_sum;
        }
        __syncthreads();
    }

    // Mean of input segments
    mean = mean / (n * gridDim.x * blockDim.x);

    // Set binary output
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}