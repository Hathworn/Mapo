#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate global index
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    // Ensure valid index within bounds
    if (i < n) {
        // Use shared memory for quick access to s[i] value (if applicable)
        float shared_s_i = s[i];  // Store s[i] in a temporary register

        // Update da if pointer is non-null
        if (da) da[i] += dc[i] * shared_s_i;

        // Update db if pointer is non-null
        if (db) db[i] += dc[i] * (1 - shared_s_i);

        // Update ds with the difference between a[i] and b[i]
        ds[i] += dc[i] * (a[i] - b[i]);
    }
}