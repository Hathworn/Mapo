#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate global thread index with grid stride loop
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = gridDim.x * blockDim.x; // Calculate total number of threads in the grid
    for(; i < n; i += gridSize) {
        c[i] += a[i] * b[i];
    }
}