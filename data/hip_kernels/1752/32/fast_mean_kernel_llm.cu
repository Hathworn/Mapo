#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Use a single loop for clearer work distribution
    // Optimize access by avoiding redundant condition checks
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += x[index];
        }
    }

    __syncthreads(); // Synchronize threads before reduction

    // Reduce using thread 0 only
    if (id == 0) {
        float sum = 0;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        mean[filter] = sum / (spatial * batch);
    }
}