#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0.0f;

    int filter = blockIdx.x;

    int j;
    for(j = 0; j < batch; ++j){
        int i;
        for(i = id; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i;
            local[id] += powf((x[index] - mean[filter]), 2.0f);
        }
    }

    __syncthreads();

    // Use parallel reduction for better performance
    for(int stride = threads / 2; stride > 0; stride >>= 1) {
        if(id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    // Use the first thread to do final computation
    if(id == 0){
        variance[filter] = local[0] / (spatial * batch - 1);
    }
}