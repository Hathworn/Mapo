#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds and if mask matches
    if (i < n) {
        float maskVal = mask[i];  // Read mask value once
        if (maskVal == mask_num)  // Compare with mask_num
            x[i] = mask_num;
    }
}