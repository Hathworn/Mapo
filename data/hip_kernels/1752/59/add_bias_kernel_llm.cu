#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    // Calculate the global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for loop

    // Iterate over all elements with stride
    for (; index < n * size * batch; index += stride) {
        int i = index % size;
        int j = (index / size) % n;
        int k = index / (size * n);

        output[(k * n + j) * size + i] += biases[j]; // Update the output
    }
}