#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Optimize index calculation by removing gridDim.y
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Use stride for iteration

    // Loop over elements with stride to cover the entire array
    for (int idx = i; idx < N; idx += stride) {
        float val = X[idx * INCX];
        if ((val * val) < (ALPHA * ALPHA)) {
            X[idx * INCX] = 0;
        }
    }
}