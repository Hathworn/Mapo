#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int k = id % c;
    int b = id / c;  // Simplify division

    int out_index = k + c * b;
    int offset = h * w * (k + b * c);
    float scale = out_delta[out_index] / (w * h);  // Calculate scale outside loop

    // Use offset and scale for efficient memory access and calculation
    for (int i = 0; i < w * h; ++i) {
        in_delta[offset + i] += scale;
    }
}