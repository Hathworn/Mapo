#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float rsqrtf_eps(float x) {
    return rsqrtf(x + .000001f); // Add small epsilon to avoid division by zero
}

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return; // Check within bounds

    int f = (index / spatial) % filters;

    // Precompute frequently used terms
    float inv_sqrt_var = rsqrtf_eps(variance[f]);
    float scale = 1.0f / (spatial * batch);

    // Optimize memory access and compute delta
    float x_minus_mean = x[index] - mean[f];
    delta[index] = delta[index] * inv_sqrt_var + variance_delta[f] * 2.f * x_minus_mean * scale + mean_delta[f] * scale;
}