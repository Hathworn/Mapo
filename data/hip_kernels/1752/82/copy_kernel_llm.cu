#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int blockSize = blockDim.x * gridDim.x * gridDim.y; // Calculate entire 1D grid size
    // Loop stride for better utilization
    for(; i < N; i += blockSize)
    {
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}