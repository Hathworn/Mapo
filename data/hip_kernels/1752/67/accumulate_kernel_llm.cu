#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Use shared memory for group sum accumulation
    __shared__ float shared_sum[1024];  // Adjust size if necessary

    shared_sum[threadIdx.x] = 0;
    for (int k = threadIdx.x; k < n; k += blockDim.x) {
        shared_sum[threadIdx.x] += x[k * groups + i];
    }

    // Synchronize threads before writing back to global memory
    __syncthreads();

    // Atomic add to aggregate results across threads
    atomicAdd(&sum[i], shared_sum[threadIdx.x]);
}