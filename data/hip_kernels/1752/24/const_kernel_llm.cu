#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate unique global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Check boundaries
    if(i < N)
    {
        // Perform computation
        X[i * INCX] = ALPHA;
    }
}