#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output) {
    float sum = 0;
    float largest = -INFINITY;
    
    // Use shared memory for parallel reduction; consider the warp size
    extern __shared__ float shared_input[];
    int tid = threadIdx.x;
    
    // Step 1: Load data into shared memory, stride-based
    for (int i = tid; i < n; i += blockDim.x) {
        shared_input[i] = input[i * stride];
        largest = max(largest, shared_input[i]);
    }
    __syncthreads();

    // Step 2: Compute softmax using shared memory
    for (int i = tid; i < n; i += blockDim.x) {
        float e = expf((shared_input[i] - largest) / temp);
        sum += e;
        output[i * stride] = e;
    }
    __syncthreads();
    
    // Step 3: Normalize the result
    for (int i = tid; i < n; i += blockDim.x) {
        output[i * stride] /= sum;
    }
}

__device__ void softmax_device(int n, float *input, float temp, float *output) {
    float sum = 0;
    float largest = -INFINITY;
    
    extern __shared__ float shared_input[];
    int tid = threadIdx.x;
    
    // Load and find largest in shared memory
    for (int i = tid; i < n; i += blockDim.x) {
        shared_input[i] = input[i];
        largest = max(largest, shared_input[i]);
    }
    __syncthreads();

    // Calculate softmax
    for (int i = tid; i < n; i += blockDim.x) {
        float e = expf((shared_input[i] - largest) / temp);
        sum += e;
        output[i] = e;
    }
    __syncthreads();
    
    // Normalize the result
    for (int i = tid; i < n; i += blockDim.x) {
        output[i] /= sum;
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output) {
    int b = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (b >= batch) return;
    
    // Utilize shared memory per block and warp
    extern __shared__ float shared_input[];
    softmax_device(n, input + b * offset, temp, shared_input, output + b * offset);
}