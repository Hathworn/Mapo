#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation
    if (idx < n) {
        float diff = truth[idx] - pred[idx];
        error[idx] = diff * diff; // Optimized error calculation
        delta[idx] = diff; // Assign delta
    }
}