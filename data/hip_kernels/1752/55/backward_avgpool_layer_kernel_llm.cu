#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    // Use blockIdx.x and threadIdx.x for indexing
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Compute k and b using efficient integer division/modulo
    int k = id % c;
    int b = id / c;

    // Precompute shared memory for division result
    float scale = 1.0f / (w * h);

    // Loop through width and height times
    int out_index = k + c * b;
    for (int i = 0; i < w * h; ++i) {
        int in_index = i + h * w * (k + b * c);
        in_delta[in_index] += out_delta[out_index] * scale;
    }
}