#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify block index calculation
    if(i < N) {
        float value = X[i*INCX];
        X[i*INCX] = fminf(ALPHA, fmaxf(-ALPHA, value)); // Compute constrained value
    }
}