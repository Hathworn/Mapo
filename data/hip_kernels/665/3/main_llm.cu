#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "restart_ants_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    struct ant *ants_d = NULL;
hipMalloc(&ants_d, XSIZE*YSIZE);
hiprandState *state_d = NULL;
hipMalloc(&state_d, XSIZE*YSIZE);
float *bestdistance_d = NULL;
hipMalloc(&bestdistance_d, XSIZE*YSIZE);
int THREADS = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
restart_ants<<<gridBlock, threadBlock>>>(ants_d,state_d,bestdistance_d,THREADS);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
restart_ants<<<gridBlock, threadBlock>>>(ants_d,state_d,bestdistance_d,THREADS);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
restart_ants<<<gridBlock, threadBlock>>>(ants_d,state_d,bestdistance_d,THREADS);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}