#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_cluster(int *cluster, float *centroid, float *B_c, float *G_c, float *R_c, int size_image, int n_threads, int K, float *Bdata, float *Gdata, float *Rdata, float *nValue)
{
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int id = threadIdx.x;
    int k = blockIdx.x;

    int size_per_thread = size_image / n_threads;
    int start = id * size_per_thread;
    int end = start + size_per_thread;

    // Initialize data
    float count = 0;
    float B = 0;
    float G = 0;
    float R = 0;
    
    nValue[tid] = 0;
    Bdata[tid] = 0;
    Gdata[tid] = 0;
    Rdata[tid] = 0;

    // Ensure id is within bounds before proceeding
    if (id >= size_image) { return; }

    // Correct partition for the last thread
    if (id == n_threads - 1)
    {
        start = (n_threads - 1) * size_per_thread;
        end = size_image;
    }

    // Calculate local B, G, R sums and count
    for (int j = start; j < end; j++)
    {
        if (cluster[j] == k)
        {
            B += B_c[j];
            G += G_c[j];
            R += R_c[j];
            count += 1;
        }
    }

    // Store local sums and count
    nValue[tid] = count;
    Bdata[tid] = B;
    Gdata[tid] = G;
    Rdata[tid] = R;

    // Synchronize before reduction
    __syncthreads();

    // Intra-block reduction
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (id < s && (tid + s) < (k + 1) * n_threads)
        {
            nValue[tid] += nValue[tid + s];
            Bdata[tid] += Bdata[tid + s];
            Gdata[tid] += Gdata[tid + s];
            Rdata[tid] += Rdata[tid + s];
        }
        __syncthreads();
    }

    // Update global centroid from reduced data
    if (id == 0)
    {
        if (nValue[tid] != 0)
        {
            centroid[k * 3 + 0] = Bdata[tid] / nValue[tid];
            centroid[k * 3 + 1] = Gdata[tid] / nValue[tid];
            centroid[k * 3 + 2] = Rdata[tid] / nValue[tid];
        }
    }
}