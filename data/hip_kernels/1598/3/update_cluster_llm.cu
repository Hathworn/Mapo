#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_cluster(int *cluster, float *centroid, float *B_c, float *G_c, float *R_c, int size_image, int n_threads, int k, float *Bdata, float *Gdata, float *Rdata, float *nValue)
{
    unsigned int tid = threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x; // Calculate stride
    int start = tid;

    float count = 0;
    float B = 0;
    float G = 0;
    float R = 0;

    // Use stride to handle oversized datasets
    for (int j = start; j < size_image; j += stride)
    {
        if (cluster[j] == k)
        {
            B += B_c[j];
            G += G_c[j];
            R += R_c[j];
            count += 1;
        }
    }

    nValue[tid] = count;
    Bdata[tid] = B;
    Gdata[tid] = G;
    Rdata[tid] = R;

    __syncthreads();

    // Optimized parallel reduction
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            nValue[tid] += nValue[tid + s];
            Bdata[tid] += Bdata[tid + s];
            Gdata[tid] += Gdata[tid + s];
            Rdata[tid] += Rdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        if (nValue[0] != 0)
        {
            centroid[k * 3 + 0] = Bdata[0] / nValue[0];
            centroid[k * 3 + 1] = Gdata[0] / nValue[0];
            centroid[k * 3 + 2] = Rdata[0] / nValue[0];
        }
    }
}