#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_gpu(void) {
    // Improved string formatting for readability and thread-based identification
    printf("Houston, we have a problem in section [thread: %d, block: %d] from Apollo 13\n", threadIdx.x, blockIdx.x);
}