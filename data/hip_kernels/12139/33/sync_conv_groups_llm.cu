#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void sync_conv_groups() {
    __syncthreads(); // Ensuring threads within the same block synchronize
}