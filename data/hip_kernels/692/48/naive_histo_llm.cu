#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void naive_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Use shared memory to accumulate histogram for current block
    extern __shared__ int shared_bins[];
    
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;
    
    // Initialize shared memory for the current block
    if (threadIdx.x < BIN_COUNT) 
    {
        shared_bins[threadIdx.x] = 0;
    }
    __syncthreads();
    
    // Accumulate histogram in shared memory
    atomicAdd(&shared_bins[myBin], 1);
    
    __syncthreads();
    
    // Merge results back to global memory
    if (threadIdx.x < BIN_COUNT) 
    {
        atomicAdd(&d_bins[threadIdx.x], shared_bins[threadIdx.x]);
    }
}