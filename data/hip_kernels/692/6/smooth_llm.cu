#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth(float * v_new, const float * v) {
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x; // Correct index calculation
    int numThreads = blockDim.x * gridDim.x;
    
    // Efficient boundary checks using min and max
    int myLeftIdx = max(0, myIdx - 1);
    int myRightIdx = min(numThreads - 1, myIdx + 1);

    // Load elements from global to local cache
    float myElt = v[myIdx];
    float myLeftElt = v[myLeftIdx];
    float myRightElt = v[myRightIdx];

    // Store the result in v_new
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
}