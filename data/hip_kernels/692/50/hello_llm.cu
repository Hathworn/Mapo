#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Calculate global thread ID for unique thread identification
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello world! I'm thread %d\n", globalThreadId);
}