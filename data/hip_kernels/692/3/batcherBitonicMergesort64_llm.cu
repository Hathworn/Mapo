#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid  = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    for (int stage = 0; stage <= 5; stage++)
    {
        for (int substage = stage; substage >= 0; substage--)
        {
            // Calculate index for comparison
            int index1 = (tid >> (substage + 1)) << (substage + 1) | (tid & ((1 << substage)-1));
            int index2 = index1 | (1 << substage);

            // Conditionally swap elements
            if (tid < index2)
            {
                bool direction = ((tid >> stage) & 1) == 0;
                if ((sdata[index1] > sdata[index2]) == direction)
                {
                    float temp = sdata[index1];
                    sdata[index1] = sdata[index2];
                    sdata[index2] = temp;
                }
            }
            __syncthreads(); // Ensure all swaps are completed
        }
    }

    d_out[tid] = sdata[tid];
}