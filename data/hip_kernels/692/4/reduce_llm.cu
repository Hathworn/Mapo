#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int* s) {
  // Use warp-level intrinsic to sum values within a warp
  s[threadIdx.x] = p;
  __syncthreads();

  for (int offset = warpSize / 2; offset > 0; offset /= 2) {
    if (threadIdx.x < offset) {
      s[threadIdx.x] += s[threadIdx.x + offset];
    }
    __syncthreads();
  }

  // Return the sum stored at the first position
  return s[0];
}

__global__ void reduce(unsigned int* d_out_shared, const unsigned int* d_in) {
  extern __shared__ unsigned int s[];
  int t = threadIdx.x;
  int p = d_in[t];
  
  // Perform shared memory reduction
  unsigned int sr = shared_reduce(p, s);
  
  // Store the result from thread 0
  if (t == 0) {
    *d_out_shared = sr;
  }
}