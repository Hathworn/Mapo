#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;
    
    // Load input into shared memory for fast access
    extern __shared__ float sdata[];
    sdata[tid] = d_in[myId];
    __syncthreads();
    
    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Ensure all threads have completed this stage
    }
    
    // Write result for this block back to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}