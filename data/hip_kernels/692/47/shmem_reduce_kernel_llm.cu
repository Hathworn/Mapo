#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shmem_reduce_kernel(float * d_out, const float * d_in) {
    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // Load shared mem from global mem
    sdata[tid] = d_in[myId];
    __syncthreads();  // Ensure entire block is loaded

    // Perform reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        // Unrolling the last warp for better performance
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();  // Ensure all adds at one stage are done
    }

    // Only thread 0 writes result for this block back to global mem
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}