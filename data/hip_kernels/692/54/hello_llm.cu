#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello() {
    // Optimize by using shared memory initialization
    __shared__ char message[50];
    if (threadIdx.x == 0) {
        snprintf(message, sizeof(message), "Hello world! I'm a thread in block %d\n", blockIdx.x);
    }
    __syncthreads(); // Ensure all threads see the initialized message
    printf("%s", message); // Each thread prints the same message
}