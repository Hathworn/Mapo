#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    // Calculate normalization constant outside of loop
    const float normalization_constant = 1.f / d_input_cdf[n - 1];
    
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within the bounds before calculation
    if (global_index_1d < n) 
    {
        // Direct computation and assignment
        d_output_cdf[global_index_1d] = d_input_cdf[global_index_1d] * normalization_constant;
    }
}