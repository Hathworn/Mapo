#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bigstencil(int* in, int* out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if i is within bounds to avoid out-of-bound access
    if (i < gridDim.x * blockDim.x) {
        out[i] = in[i] + 2;
    }
}