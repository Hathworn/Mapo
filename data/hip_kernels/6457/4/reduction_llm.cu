#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];
    
    int tid = threadIdx.x;
    int index = tid + blockIdx.x * blockDim.x;
    
    tmp[tid] = input[index];
    
    __syncthreads();
    
    // Unrolling the loop for better performance
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            tmp[tid] += tmp[tid + stride];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        output[blockIdx.x] = tmp[0];
    }
}