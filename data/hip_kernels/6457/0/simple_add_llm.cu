#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_add(float* __restrict__ a, const float* __restrict__ b, int n)
{
    // Utilize built-in variables for optimized memory handling
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bounds to prevent memory overflow
    if(i < n) {
        // Perform addition operation
        a[i] += b[i];
    }
}