#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    // Cache vector_in into shared memory for faster access
    extern __shared__ double shared_vector_in[];
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    if (threadIdx.x < dim_mn) {
        shared_vector_in[threadIdx.x] = vector_in[threadIdx.x];
    }
    __syncthreads(); // Ensure all threads have loaded vector_in into shared memory

    // Perform matrix-vector multiplication
    double out = 0.0;
    if (i < dim_mn) {
        for (long j = 0; j < dim_mn; j++) {
            out += matrix[i * dim_mn + j] * shared_vector_in[j];
        }
        vector_out[i] = out;
    }
}