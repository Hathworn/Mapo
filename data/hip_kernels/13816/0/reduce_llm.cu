#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce(int *in, int *out, int N) {
    extern __shared__ int sdata[]; // Shared memory for block reduction
    int tid = threadIdx.x;

    // Each thread calculates partial sum
    int sum = 0;
    for (int i = blockIdx.x * blockDim.x + tid; i < N; i += blockDim.x * gridDim.x) {
        sum += in[i];
    }

    // Store partial sum in shared memory
    sdata[tid] = sum;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Only one thread per block writes to global output
    if (tid == 0) {
        atomicAdd(out, sdata[0]);
    }
}