#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vector_add(double const *A_dev, double const *B_dev, double *C_dev, int const N) {
    // Calculate global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimize boundary check and assignment
    if (i < N) {
        C_dev[i] = A_dev[i] + B_dev[i];
    }
}