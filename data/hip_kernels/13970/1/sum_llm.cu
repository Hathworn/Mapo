#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(int *dest, int a, int b) {
    // Use blockIdx and threadIdx to ensure it's optimized for multiple threads
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) { // Ensure only first thread performs addition
        *dest = a + b;
    }
}