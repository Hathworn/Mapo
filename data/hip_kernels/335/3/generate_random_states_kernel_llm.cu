#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_random_states_kernel(unsigned int seed, hiprandState_t* d_states, size_t total_number) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Check if within bounds and initialize state if true
    if (idx < total_number) {
        hiprand_init(seed, idx, 0, &d_states[idx]);
    }
    // No need for __syncthreads() as it is not required for independent states
}