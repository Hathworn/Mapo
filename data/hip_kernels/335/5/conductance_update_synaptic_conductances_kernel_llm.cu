#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conductance_update_synaptic_conductances_kernel(float timestep, float * d_synaptic_conductances_g, float * d_synaptic_efficacies_or_weights, float * d_time_of_last_spike_to_reach_synapse, float * d_biological_conductance_scaling_constants_lambda, int total_number_of_synapses, float current_time_in_seconds, float * d_decay_terms_tau_g) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Cache stride computation
    while (idx < total_number_of_synapses) {
        float synaptic_conductance_g = d_synaptic_conductances_g[idx];
        float decay_term = 1.0 - (timestep / d_decay_terms_tau_g[idx]); // Cache decay computation
        float new_conductance = decay_term * synaptic_conductance_g;

        if (d_time_of_last_spike_to_reach_synapse[idx] == current_time_in_seconds) {
            // Combine multipliers into a single operation for new conductance update
            new_conductance += timestep * d_synaptic_efficacies_or_weights[idx] * d_biological_conductance_scaling_constants_lambda[idx];
        }

        // Only write to global memory if there's a change
        if (synaptic_conductance_g != new_conductance) {
            d_synaptic_conductances_g[idx] = new_conductance;
        }

        idx += stride;
    }
}