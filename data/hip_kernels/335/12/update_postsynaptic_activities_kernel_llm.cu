#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_postsynaptic_activities_kernel(float timestep, size_t total_number_of_neurons, float *d_recent_postsynaptic_activities_D, float *d_last_spike_time_of_each_neuron, float current_time_in_seconds, float decay_term_tau_D, float model_parameter_alpha_D) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll the loop for better performance.
    for (int i = idx; i < total_number_of_neurons; i += blockDim.x * gridDim.x) {
        float recent_postsynaptic_activity_D = d_recent_postsynaptic_activities_D[i];
        
        // Combine calculations to reduce operations.
        float new_recent_postsynaptic_activity_D = recent_postsynaptic_activity_D * (1 - (timestep / decay_term_tau_D));
        
        if (d_last_spike_time_of_each_neuron[i] == current_time_in_seconds) {
            new_recent_postsynaptic_activity_D += timestep * model_parameter_alpha_D * (1 - recent_postsynaptic_activity_D);
        }
        
        d_recent_postsynaptic_activities_D[i] = new_recent_postsynaptic_activity_D;
    }
}