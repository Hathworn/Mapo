#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void reset_states_u_after_spikes_kernel(float *d_states_u, float *d_param_d, float* d_last_spike_time_of_each_neuron, float current_time_in_seconds, size_t total_number_of_neurons) {
    // Calculate index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Iterate over neurons efficiently
    for (; idx < total_number_of_neurons; idx += blockDim.x * gridDim.x) {
        // Check for spike time match and update state
        if (d_last_spike_time_of_each_neuron[idx] == current_time_in_seconds) {
            d_states_u[idx] += d_param_d[idx]; // Perform state update
        }
    }
}