#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void check_for_generator_spikes_kernel(int *d_neuron_ids_for_stimulus, float *d_spike_times_for_stimulus, float* d_last_spike_time_of_each_neuron, unsigned char* d_bitarray_of_neuron_spikes, int bitarray_length, int bitarray_maximum_axonal_delay_in_timesteps, float current_time_in_seconds, float timestep, size_t number_of_spikes_in_stimulus, bool high_fidelity_spike_flag) {

    // Calculate thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    
    while (idx < number_of_spikes_in_stimulus) {
        float spike_time_difference = fabs(current_time_in_seconds - d_spike_times_for_stimulus[idx]);
        
        if (spike_time_difference < 0.5f * timestep) {
            d_last_spike_time_of_each_neuron[d_neuron_ids_for_stimulus[idx]] = current_time_in_seconds;
        }
        
        if (high_fidelity_spike_flag) {
            // Calculate start of bit storage for this neuron
            int neuron_id_spike_store_start = d_neuron_ids_for_stimulus[idx] * bitarray_length;
            
            // Compute offset for current timestep
            int offset_index = (int)(roundf(current_time_in_seconds / timestep)) % bitarray_maximum_axonal_delay_in_timesteps;
            int offset_byte = offset_index >> 3;  // Equivalent to offset_index / 8
            int offset_bit_pos = offset_index & 7; // Equivalent to offset_index % 8
            
            // Get current byte and update required bit
            unsigned char* byte_ptr = &d_bitarray_of_neuron_spikes[neuron_id_spike_store_start + offset_byte];
            if (spike_time_difference < 0.5f * timestep) {
                atomicOr(byte_ptr, (1 << offset_bit_pos));
            } else {
                atomicAnd(byte_ptr, ~(1 << offset_bit_pos));
            }
        }
        
        idx += stride;
    }
    
    __syncthreads();
}