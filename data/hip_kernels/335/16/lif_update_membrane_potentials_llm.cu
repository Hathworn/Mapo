#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lif_update_membrane_potentials(float *d_membrane_potentials_v, float *d_membrane_resistances_R, float *d_membrane_time_constants_tau_m, float *d_resting_potentials, float* d_current_injections, float timestep, size_t total_number_of_neurons) {
    // Calculate global thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    // Process multiple neurons per thread to improve occupancy
    int stride = blockDim.x * gridDim.x;
    
    for (int neuron_idx = idx; neuron_idx < total_number_of_neurons; neuron_idx += stride) {
        float equation_constant = timestep / d_membrane_time_constants_tau_m[neuron_idx];
        float membrane_potential_Vi = d_membrane_potentials_v[neuron_idx];
        float current_injection_Ii = d_current_injections[neuron_idx];
        float resting_potential_V0 = d_resting_potentials[neuron_idx];
        float temp_membrane_resistance_R = d_membrane_resistances_R[neuron_idx];
        
        // Compute new membrane potential
        float new_membrane_potential = equation_constant * (resting_potential_V0 + temp_membrane_resistance_R * current_injection_Ii) + (1 - equation_constant) * membrane_potential_Vi;

        d_membrane_potentials_v[neuron_idx] = new_membrane_potential;
    }
    // Remove __syncthreads() as it's unnecessary here
}