#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void izhikevich_update_membrane_potentials_kernel(float *d_membrane_potentials_v, float *d_states_u, float *d_param_a, float *d_param_b, float* d_current_injections, float timestep, size_t total_number_of_neurons) {

    // Precompute factor outside loop
    float eqtimestep = timestep * 1000.0f;

    // Calculate global thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Efficiently iterate over neurons using stride loop pattern
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < total_number_of_neurons; i += stride) {
        // Update membrane potentials using Izhikevich equations
        float v_update = 0.04f * d_membrane_potentials_v[i] * d_membrane_potentials_v[i]
                         + 5.0f * d_membrane_potentials_v[i]
                         + 140.0f
                         - d_states_u[i]
                         + d_current_injections[i];

        // Atomic operations to avoid race conditions
        d_membrane_potentials_v[i] += eqtimestep * v_update;
        d_states_u[i] += eqtimestep * (d_param_a[i] * (d_param_b[i] * d_membrane_potentials_v[i] - d_states_u[i]));
    }

    // Synchronization not needed within kernel scope for this use case
}