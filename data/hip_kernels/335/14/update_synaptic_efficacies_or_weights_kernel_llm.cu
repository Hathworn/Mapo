#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_synaptic_efficacies_or_weights_kernel(float * d_recent_presynaptic_activities_C, float * d_recent_postsynaptic_activities_D, int* d_postsynaptic_neuron_indices, float* d_synaptic_efficacies_or_weights, float current_time_in_seconds, float * d_time_of_last_spike_to_reach_synapse, float * d_last_spike_time_of_each_neuron, bool* d_stdp, size_t total_number_of_synapses, float learning_rate_rho) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Use a more efficient loop by handling condition checks in fewer operations
    for (; idx < total_number_of_synapses; idx += blockDim.x * gridDim.x) {
        if (d_stdp[idx]) {
            float synaptic_efficacy_delta_g = d_synaptic_efficacies_or_weights[idx];
            float new_synaptic_efficacy = synaptic_efficacy_delta_g;
            float new_componet = 0.0f;
            
            int postsynaptic_neuron_index = d_postsynaptic_neuron_indices[idx];

            // Combine condition checks to reduce branching
            if (d_last_spike_time_of_each_neuron[postsynaptic_neuron_index] == current_time_in_seconds) {
                new_componet += (1.0f - synaptic_efficacy_delta_g) * d_recent_presynaptic_activities_C[idx];
            }

            if (d_time_of_last_spike_to_reach_synapse[idx] == current_time_in_seconds) {
                new_componet -= synaptic_efficacy_delta_g * d_recent_postsynaptic_activities_D[postsynaptic_neuron_index];
            }

            // Use a direct computation to update synaptic efficacy
            if (new_componet != 0.0f) {
                new_synaptic_efficacy += learning_rate_rho * new_componet;
                // Clamp the updated synaptic efficacy between 0 and 1
                new_synaptic_efficacy = fminf(fmaxf(new_synaptic_efficacy, 0.0f), 1.0f);

                // Update synaptic efficacies
                if (synaptic_efficacy_delta_g != new_synaptic_efficacy) {
                    d_synaptic_efficacies_or_weights[idx] = new_synaptic_efficacy;
                }
            }
        }
    }
}