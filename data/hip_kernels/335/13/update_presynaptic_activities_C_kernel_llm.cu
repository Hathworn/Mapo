#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_presynaptic_activities_C_kernel(float* d_recent_presynaptic_activities_C, float* d_time_of_last_spike_to_reach_synapse, bool* d_stdp, float timestep, float current_time_in_seconds, size_t total_number_of_synapses, float synaptic_neurotransmitter_concentration_alpha_C, float decay_term_tau_C) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Loop unrolling for better occupancy
    for (size_t stride = blockDim.x * gridDim.x; idx < total_number_of_synapses; idx += stride) {
        if (d_stdp[idx]) {
            float recent_presynaptic_activity_C = d_recent_presynaptic_activities_C[idx];
            float decay_factor = (1 - (timestep / decay_term_tau_C));
            float new_recent_presynaptic_activity_C = decay_factor * recent_presynaptic_activity_C;

            // Consolidated condition check to reduce branch divergence
            if (d_time_of_last_spike_to_reach_synapse[idx] == current_time_in_seconds) {
                new_recent_presynaptic_activity_C += timestep * synaptic_neurotransmitter_concentration_alpha_C * (1 - recent_presynaptic_activity_C);
            }

            d_recent_presynaptic_activities_C[idx] = new_recent_presynaptic_activity_C;
        }
    }
}