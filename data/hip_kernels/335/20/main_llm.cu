#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "poisson_update_membrane_potentials_kernel_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    hiprandState_t *d_states = NULL;
hipMalloc(&d_states, XSIZE*YSIZE);
float *d_rates = NULL;
hipMalloc(&d_rates, XSIZE*YSIZE);
float *d_membrane_potentials_v = NULL;
hipMalloc(&d_membrane_potentials_v, XSIZE*YSIZE);
float timestep = 1;
float *d_thresholds_for_action_potential_spikes = NULL;
hipMalloc(&d_thresholds_for_action_potential_spikes, XSIZE*YSIZE);
size_t total_number_of_input_neurons = 1;
int current_stimulus_index = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
poisson_update_membrane_potentials_kernel<<<gridBlock, threadBlock>>>(d_states,d_rates,d_membrane_potentials_v,timestep,d_thresholds_for_action_potential_spikes,total_number_of_input_neurons,current_stimulus_index);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
poisson_update_membrane_potentials_kernel<<<gridBlock, threadBlock>>>(d_states,d_rates,d_membrane_potentials_v,timestep,d_thresholds_for_action_potential_spikes,total_number_of_input_neurons,current_stimulus_index);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
poisson_update_membrane_potentials_kernel<<<gridBlock, threadBlock>>>(d_states,d_rates,d_membrane_potentials_v,timestep,d_thresholds_for_action_potential_spikes,total_number_of_input_neurons,current_stimulus_index);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}