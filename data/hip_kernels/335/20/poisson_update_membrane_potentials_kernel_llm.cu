#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poisson_update_membrane_potentials_kernel(hiprandState_t* d_states, float *d_rates, float *d_membrane_potentials_v, float timestep, float *d_thresholds_for_action_potential_spikes, size_t total_number_of_input_neurons, int current_stimulus_index) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    while (idx < total_number_of_input_neurons) {
        
        // Calculate rate index directly from idx
        int rate_index = total_number_of_input_neurons * current_stimulus_index + idx;

        float rate = d_rates[rate_index];

        if (rate > 0.1) {

            // Generate random float using a separate seed for each thread
            float random_float = hiprand_uniform(&d_states[idx]);

            // Check if random number is less than rate * timestep
            if (random_float < (rate * timestep)) {

                // Update membrane potential if condition is met
                d_membrane_potentials_v[idx] = d_thresholds_for_action_potential_spikes[idx] + 0.02;
            }
        }

        idx += blockDim.x * gridDim.x;
    }
}