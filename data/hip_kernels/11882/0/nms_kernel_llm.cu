#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DIVUP(m,n)		((m)/(n)+((m)%(n)>0))
int const threadsPerBlock = (sizeof(unsigned long long) * 8);

/*
* Device code
*/
__device__ inline float devIoU(float const * const a, float const * const b)
{
    float left = max(a[0], b[0]), right = min(a[2], b[2]);
    float top = max(a[1], b[1]), bottom = min(a[3], b[3]);
    float width = max(right - left + 1, 0.f), height = max(bottom - top + 1, 0.f);
    float interS = width * height;
    float Sa = (a[2] - a[0] + 1) * (a[3] - a[1] + 1);
    float Sb = (b[2] - b[0] + 1) * (b[3] - b[1] + 1);
    return interS / (Sa + Sb - interS);
}

__global__ void nms_kernel(const int n_boxes, const float nms_overlap_thres, const float *dev_boxes, unsigned long long *dev_mask)
{
    const int row_start = blockIdx.y, col_start = blockIdx.x;
    const int row_size = min(n_boxes - row_start * threadsPerBlock, threadsPerBlock), col_size = min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

    // Use shared memory efficiently by loading the necessary box data only
    __shared__ float block_boxes[threadsPerBlock * 5];
    if (threadIdx.x < col_size)
    {
        const int box_offset = (threadsPerBlock * col_start + threadIdx.x) * 5;
        #pragma unroll
        for (int j = 0; j < 5; j++)
        {
            block_boxes[threadIdx.x * 5 + j] = dev_boxes[box_offset + j];
        }
    }
    __syncthreads();

    if (threadIdx.x < row_size)
    {
        const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
        const float *cur_box = dev_boxes + cur_box_idx * 5;
        unsigned long long t = 0;
        int start = row_start == col_start ? threadIdx.x + 1 : 0;
        for (int i = start; i < col_size; i++)
        {
            if (devIoU(cur_box, block_boxes + i * 5) > nms_overlap_thres)
            {
                t |= 1ULL << i;
            }
        }
        const int col_blocks = DIVUP(n_boxes, threadsPerBlock);
        dev_mask[cur_box_idx * col_blocks + col_start] = t;
    }
}