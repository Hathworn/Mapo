#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate index with fewer registers
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use branch-less condition with ternary operator
    if (ptidx < npoints)
    {
        // Load data from global memory once, optimization with register reuse
        short2 loc = loc_[ptidx];
        
        // Optimize memory accesses by coalescing
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}