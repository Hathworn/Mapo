#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Precompute and cache reused variables
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    int base_tx = (int) px;
    int base_ty = (int) py;

    float value = src[image_row_offset + j];

    // Consolidated bounds checking
    int positions[4][2] = {
        {0, 0},
        {-1, 0},
        {-1, -1},
        {0, -1}
    };

    float weights[4] = {
        dx * dy,
        (1.0f - dx) * dy,
        (1.0f - dx) * (1.0f - dy),
        dx * (1.0f - dy)
    };

    #pragma unroll
    for (int k = 0; k < 4; k++) {
        int tx = base_tx + positions[k][0];
        int ty = base_ty + positions[k][1];
        
        if (!(tx >= w || tx < 0 || ty >= h || ty < 0)) {
            float weight = weights[k];
            _atomicAdd(dst + ty * image_stride + tx, value * weight);
            _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
        }
    }
}