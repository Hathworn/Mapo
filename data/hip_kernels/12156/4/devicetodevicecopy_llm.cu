#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void devicetodevicecopy(double *dphi, double *dpsix, double *dpsiy, double *mphi, double *mpsix, double *mpsiy, unsigned int nx, unsigned int TileSize)
{
    // Calculate global thread indices
    unsigned int index_x = blockIdx.x * TileSize + threadIdx.x;
    unsigned int index_y = blockIdx.y * TileSize + threadIdx.y;

    // Derive global index to write
    unsigned int indexToWrite = index_y * nx + index_x;

    // Avoid out-of-bound memory access
    if (index_x < nx && index_y * nx < nx * nx) {
        mphi[indexToWrite] = dphi[indexToWrite];
        mpsix[indexToWrite] = dpsix[indexToWrite];
        mpsiy[indexToWrite] = dpsiy[indexToWrite];
    }
}