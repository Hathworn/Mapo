#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int locationAlgo(double *x, double xadv, unsigned int nx)
{
    unsigned int location = 0;
    while (x[location] < xadv && location < nx)
        location++;
    if (location == 0)
        return location;
    else
        return location - 1;
}

__global__ void find_advection_point_location_cuda(double *x, double *y, double *xadv, double *yadv, unsigned int nx, unsigned int ny, unsigned int *cellx, unsigned int *celly, unsigned int *tracker, double xlim1, double xlim2, double ylim1, double ylim2, unsigned int TileSize)
{
    unsigned int index_x = blockIdx.x * TileSize + threadIdx.x;
    unsigned int index_y = blockIdx.y * TileSize + threadIdx.y;
    unsigned int indexToWrite = index_y * nx + index_x;

    // Check bounds for x and y
    bool xoutofbounds = (xadv[indexToWrite] <= xlim1 || xadv[indexToWrite] >= xlim2);
    bool youtofbounds = (yadv[indexToWrite] <= ylim1 || yadv[indexToWrite] >= ylim2);

    // Optimize branch conditions using single pass logic
    tracker[indexToWrite] = xoutofbounds * 2 + youtofbounds;

    if (tracker[indexToWrite] == 0) // in bounds
    {
        cellx[indexToWrite] = locationAlgo(x, xadv[indexToWrite], nx);
        celly[indexToWrite] = locationAlgo(y, yadv[indexToWrite], ny);
    }
    else if (tracker[indexToWrite] == 1) // y out of bounds
    {
        cellx[indexToWrite] = locationAlgo(x, xadv[indexToWrite], nx);
        celly[indexToWrite] = (yadv[indexToWrite] <= ylim1) ? 0 : ny - 2;
    }
    else if (tracker[indexToWrite] == 2) // x out of bounds
    {
        celly[indexToWrite] = locationAlgo(y, yadv[indexToWrite], ny);
        cellx[indexToWrite] = (xadv[indexToWrite] <= xlim1) ? 0 : nx - 2;
    }
    // tracker 3 means both out of bounds, no action needed
}