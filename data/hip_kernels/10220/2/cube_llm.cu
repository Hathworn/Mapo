#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cube(float * d_out, float * d_in)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global thread ID
    float num = d_in[id];
    d_out[id] = num * num * num; // Compute cube and store in output
}