#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use local variables to cache block and thread indices for improved readability
    int block_id = blockIdx.x;
    int thread_id = threadIdx.x;

    // Use formatted string and separate print calls for efficiency
    printf("Hello world! ");
    printf("blockid: %d\n", block_id);
    printf("threadid: %d\n", thread_id);
}