#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    x = ii / d1;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    x = ii / d1;
    w /= scale_factor;
    z /= scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x;
    ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
    if (ii >= no_elements) return;

    // Optimize memory access using shared memory
    extern __shared__ float shared_data[];
    int thread_id = threadIdx.y * blockDim.x + threadIdx.x;

    // Load gradOutput_data into shared memory
    if (thread_id < scale_factor * scale_factor) {
        for (int i = 0; i < scale_factor; i++) {
            for (int j = 0; j < scale_factor; j++) {
                int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
                shared_data[thread_id] = gradOutput_data[ipidx];
            }
        }
    }
    __syncthreads();  // Ensure all threads have loaded their portion

    // Accumulate results
    if (thread_id < scale_factor * scale_factor) {
        for (int i = 0; i < scale_factor; i++) {
            for (int j = 0; j < scale_factor; j++) {
                gradInput_data[ii] += shared_data[thread_id];
            }
        }
    }
}