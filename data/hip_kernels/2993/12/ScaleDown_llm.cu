#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ScaleDown(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
    __shared__ float inrow[SCALEDOWN_W+4];
    __shared__ float brow[5*(SCALEDOWN_W/2)];
    __shared__ int yRead[SCALEDOWN_H+4];
    __shared__ int yWrite[SCALEDOWN_H+4];
    #define dx2 (SCALEDOWN_W/2)
    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * SCALEDOWN_W;
    const int yStart = blockIdx.y * SCALEDOWN_H;
    const int xWrite = xStart / 2 + tx;
    float k0 = d_ScaleDownKernel[0];
    float k1 = d_ScaleDownKernel[1];
    float k2 = d_ScaleDownKernel[2];

    if (tx < SCALEDOWN_H + 4) {
        int y = yStart + tx - 2;
        y = max(0, min(y, height - 1));  // Use `max` and `min` for bounds check
        yRead[tx] = y * pitch;
        yWrite[tx] = (yStart + tx - 4) / 2 * newpitch;
    }
    __syncthreads();

    int maxtx = min(dx2, width / 2 - xStart / 2);

    for (int dy = 0; dy < SCALEDOWN_H + 4; dy += 5) {
        int xRead = xStart + tx - 2;
        xRead = max(0, min(xRead, width - 1));  // Use `max` and `min` for bounds check

        inrow[tx] = d_Data[yRead[dy] + xRead];
        __syncthreads();

        if (tx < maxtx) {
            brow[tx + 3 * dx2] = k0 * (inrow[tx * 2] + inrow[tx * 2 + 4]) + k1 * (inrow[tx * 2 + 1] + inrow[tx * 2 + 3]) + k2 * inrow[tx * 2 + 2];
            if (dy >= 4 && !(dy & 1)) {
                d_Result[yWrite[dy] + xWrite] = k2 * brow[tx + 2 * dx2] + k0 * (brow[tx] + brow[tx + 3 * dx2]) + k1 * (brow[tx + 1 * dx2] + brow[tx + 2 * dx2 + dx2]);
            }
        }
        __syncthreads();
    }
}
```
