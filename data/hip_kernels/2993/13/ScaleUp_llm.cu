#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ScaleUp(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch)
{
    #undef BW
    #undef BH
    #define BW (SCALEUP_W/2 + 2)
    #define BH (SCALEUP_H/2 + 2)
    __shared__ float buffer[BW*BH];

    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    const int bx = blockIdx.x * (SCALEUP_W / 2);
    const int by = blockIdx.y * (SCALEUP_H / 2);

    int x = min(max(bx + tx - 1, 0), width - 1);
    int y = min(max(by + ty - 1, 0), height - 1);

    // Load data into shared memory
    if (tx < BW && ty < BH) {
        buffer[ty * BW + tx] = d_Data[y * pitch + x];
    }
    __syncthreads();

    // Calculate output position
    x = bx * 2 + tx;
    y = by * 2 + ty;

    if (x < 2 * width && y < 2 * height) {
        int localX = (tx + 1) / 2;
        int localY = (ty + 1) / 2;
        int index = localY * BW + localX;
        float wx = 0.25f + (tx & 1) * 0.50f;
        float wy = 0.25f + (ty & 1) * 0.50f;

        // Compute interpolated value
        d_Result[y * newpitch + x] = 
            wy * (wx * buffer[index] + (1.0f - wx) * buffer[index + 1]) +
            (1.0f - wy) * (wx * buffer[index + BW] + (1.0f - wx) * buffer[index + BW + 1]);
    }
}