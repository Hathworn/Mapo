#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void grayscale(unsigned char *rgb, unsigned char *g, std::size_t cols, std::size_t rows) {
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < cols && j < rows) {
        // Precompute index base for pixel access
        auto index = 3 * (j * cols + i);
        // Use registers for intermediate results to improve performance
        unsigned char r = rgb[index];
        unsigned char b = rgb[index + 1];
        unsigned char g_ = rgb[index + 2];
        g[j * cols + i] = (307 * r + 604 * b + 113 * g_) / 1024;
    }
}