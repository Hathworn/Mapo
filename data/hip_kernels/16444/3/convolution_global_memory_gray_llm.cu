#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution_global_memory_gray(unsigned char *N, float *M, unsigned char* g, std::size_t cols, std::size_t rows, std::size_t mask_size) {
    int paddingSize = (mask_size - 1) / 2;
    unsigned int paddedH = cols + 2 * paddingSize;
    unsigned int paddedW = rows + 2 * paddingSize;

    int i = blockIdx.x * blockDim.x + threadIdx.x + paddingSize;
    int j = blockIdx.y * blockDim.y + threadIdx.y + paddingSize;

    // Use shared memory for tile caching
    extern __shared__ unsigned char sharedMem[];
    int localIdx = threadIdx.y * blockDim.x + threadIdx.x;
    sharedMem[localIdx] = 0;

    if ((j >= paddingSize) && (j < paddedW - paddingSize) && (i >= paddingSize) && (i < paddedH - paddingSize)) {
        unsigned int oPixelPos = (j - paddingSize) * cols + (i - paddingSize);
        float sum = 0.0f;  // Use a local variable to accumulate results
      
        for (int k = -paddingSize; k <= paddingSize; k++) {
            for (int l = -paddingSize; l <= paddingSize; l++) {
                unsigned int iPixelPos = (j + l) * cols + (i + k);
                unsigned int coefPos = (k + paddingSize) * mask_size + (l + paddingSize);
                sum += N[iPixelPos] * M[coefPos];
            }
        }
        g[oPixelPos] = static_cast<unsigned char>(sum);  // Assign the accumulated result
    }
}