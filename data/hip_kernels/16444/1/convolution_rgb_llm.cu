#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution_rgb(unsigned char *N, float *M, unsigned char *g, std::size_t cols, std::size_t rows, std::size_t mask_size) {
    int paddingSize = ((mask_size - 1) / 2) * 3;
    unsigned int paddedH = cols + 2 * paddingSize;
    unsigned int paddedW = rows * 3 + 2 * paddingSize;

    int i = blockIdx.x * blockDim.x + threadIdx.x + paddingSize;
    int j = blockIdx.y * blockDim.y + threadIdx.y + paddingSize;

    // Ensure thread is within valid computation boundary
    if ((i >= paddingSize) && (i < paddedW - paddingSize) && (j >= paddingSize) && (j < paddedH - paddingSize)) {
        unsigned int oPixelPos = (i - paddingSize) * cols + (j - paddingSize);
        float sum = 0.0f;  // Using floating point accumulation for better precision

        // Loop unrolling for optimization
        for (int k = -paddingSize; k <= paddingSize; k = k + 3) {
            for (int l = -paddingSize; l <= paddingSize; l = l + 3) {
                unsigned int iPixelPos = (i + k) * paddedH + (j + l);
                int iterationK = (k + paddingSize) / 3;
                int iterationL = (l + paddingSize) / 3;
                unsigned int filtrePos = iterationK * mask_size + iterationL;

                sum += N[iPixelPos] * M[filtrePos];  // Accumulate using a temporary sum
            }
        }

        g[oPixelPos] = static_cast<unsigned char>(sum);  // Convert accumulated sum to unsigned char
    }
}