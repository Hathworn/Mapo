#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)

__global__ void sayHi()
{
    // Use thread block and thread index for potential use or future extensibility
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0)  // Limit printf usage to a single thread to avoid cluttered output
    {
        printf("Cuda Kernel Hello World.\n");
    }
}