#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define LEN 1024
#define SIZE LEN * sizeof(float)
#define ITER 1024*1024

__global__ void Inc1(float *Ad, float *Bd) {
    // Use shared memory for performance improvement
    __shared__ float sharedA[1024];
    __shared__ float sharedB[1024];

    int tx = threadIdx.x + blockIdx.x * blockDim.x;

    if (tx < 1) {
        float ad_val = Ad[tx];

        for (int i = 0; i < ITER; i++) {
            ad_val += 1.0f;

            // Split the loop into smaller pieces to reduce control overhead
            #pragma unroll
            for (int j = 0; j < 256; j++) {
                sharedB[tx] = ad_val;
            }
        }

        Ad[tx] = ad_val;
        Bd[tx] = sharedB[tx];
    }
}