#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelA(float* r, float* x, float* y, float* z, int size)
{
    // Use blockIdx.x and blockDim.x for optimized access patterns.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure idx is within array bounds.
    if (idx < size) {
        r[idx] = x[idx] * y[idx] + z[idx];
    }
}