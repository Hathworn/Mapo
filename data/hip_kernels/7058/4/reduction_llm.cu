#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction(int * in, int * out) {
    int globalid = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int s_array[BLOCK_DIM];

    // Load elements into shared memory
    s_array[threadIdx.x] = in[globalid];
    __syncthreads();

    // Perform reduction using shared memory with stride loop unrolling
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            s_array[threadIdx.x] += s_array[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write the block's result
    if (threadIdx.x == 0) {
        out[blockIdx.x] = s_array[0];
    }
}