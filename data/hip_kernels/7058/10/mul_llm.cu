#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul(int *A, int *B, int *C) {
    // Use shared memory for tiles of A and B to reduce global memory accesses
    __shared__ int tileA[TILE_SIZE][TILE_SIZE];
    __shared__ int tileB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    int temp = 0;

    for (int m = 0; m < (N + TILE_SIZE - 1) / TILE_SIZE; ++m) {
        if (row < N && m * TILE_SIZE + threadIdx.x < N) {
            tileA[threadIdx.y][threadIdx.x] = A[row * N + m * TILE_SIZE + threadIdx.x];
        } else {
            tileA[threadIdx.y][threadIdx.x] = 0;
        }
        if (col < N && m * TILE_SIZE + threadIdx.y < N) {
            tileB[threadIdx.y][threadIdx.x] = B[(m * TILE_SIZE + threadIdx.y) * N + col];
        } else {
            tileB[threadIdx.y][threadIdx.x] = 0;
        }

        __syncthreads();

        for (int k = 0; k < TILE_SIZE; ++k) {
            temp += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < N && col < N) {
        C[row * N + col] = temp;
    }
}