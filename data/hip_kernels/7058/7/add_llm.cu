#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int * A, int * B, int * C){
    // Calculate global thread index
    int thread = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure we do not go out of bounds
    if(thread < gridDim.x * blockDim.x) {
        C[thread] = A[thread] + B[thread];
    }
}