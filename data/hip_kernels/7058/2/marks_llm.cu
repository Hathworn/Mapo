#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void marks(float *media, int *final){
    int thread = blockIdx.x * blockDim.x + threadIdx.x;
    float value = media[thread];
    int intValue = (int)value;
    
    // Use one conditional statement and simplify calculations
    if (value == intValue) {
        final[thread] = intValue;
    } else if (value > 4 && value < 5) {
        final[thread] = 4;
    } else if (value > 9) {
        final[thread] = 9;
    } else {
        final[thread] = intValue + 1;
    }
}