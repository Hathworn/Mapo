#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudahello() {
    // Calculate global thread index for detailed identification
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hola Mundo! Soy el hilo global %d (hilo local %d del bloque %d)\n", globalThreadId, threadIdx.x, blockIdx.x);
}