#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction(int *in, int *out) {
    int globalid = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int s_array[]; // Use dynamic shared memory

    // Load input into shared memory
    s_array[threadIdx.x] = in[globalid];
    __syncthreads();

    // Perform reduction within the block
    for (unsigned int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            s_array[threadIdx.x] += s_array[threadIdx.x + i];
        }
        __syncthreads();
    }

    // Write the result of this block to output
    if (threadIdx.x == 0) {
        out[blockIdx.x] = s_array[0];
    }
}