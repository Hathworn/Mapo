#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cube(double* d_out, const double* __restrict__ d_in)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Use global index
    double f = d_in[idx];
    d_out[idx] = f * f * f; // Maintain load/store efficiency
}