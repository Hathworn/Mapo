#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global thread index for scalability
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread index is within bounds to handle any excess threads
    if (i < size)
    {
        c[i] = a[i] + b[i];
    }
}