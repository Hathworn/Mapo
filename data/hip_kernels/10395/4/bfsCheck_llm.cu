#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bfsCheck(bool *d_graph_mask, bool *d_updating_graph_mask, bool *d_graph_visited, int no_of_nodes, bool *stop)
{
    __shared__ bool local_stop; // Use shared memory for stop flag
    if (threadIdx.x == 0) local_stop = false; // Initialize shared stop flag
    __syncthreads();

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < no_of_nodes) {
        if (d_updating_graph_mask[tid]) {
            d_graph_mask[tid] = true;
            d_graph_visited[tid] = true;
            local_stop = true; // Set shared stop flag
            d_updating_graph_mask[tid] = false;
        }
    }
    __syncthreads();
    if (threadIdx.x == 0 && local_stop) *stop = true; // Update global stop flag
}