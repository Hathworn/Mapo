#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory and memory coalescing for improved performance
    if (tid < N) {
        float value = indata[tid];
        ingrad[tid] = value > 0 ? outgrad[tid] : 0.0f;
    }
}