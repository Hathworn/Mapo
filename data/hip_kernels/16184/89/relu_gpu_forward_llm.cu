#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate the stride for loop unrolling
    for (int i = tid; i < N; i += stride) { // Unroll with stride to leverage memory access
        float value = in[i];
        out[i] = value > 0 ? value : 0;
    }
}