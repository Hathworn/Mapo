#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    __shared__ float uni;
    
    // Load uni value from global to shared memory by the first thread in the block
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];
    
    __syncthreads(); // Ensure that the shared memory is updated before proceeding

    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x;

    // Optimize by reducing condition check in g_data[address + blockDim.x] update
    if (address < n) {
        g_data[address] += uni;
        if (address + blockDim.x < n)
            g_data[address + blockDim.x] += uni;
    }
}