#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void double_value(double *x, double *y)
{
    // Calculate global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within array bounds
    if (idx < N) // Replace N with the correct array size
    {
        // Perform computation
        y[idx] = 2.0 * x[idx];
    }
}