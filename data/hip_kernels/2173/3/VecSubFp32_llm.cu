#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecSubFp32(float* __restrict__ in0, float* __restrict__ in1, float* __restrict__ out, int cnt)
{
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within bounds
    if (tid < cnt) {
        // Perform subtraction
        out[tid] = in0[tid] - in1[tid];
    }
}