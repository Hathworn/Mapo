#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAddFp32(float* __restrict__ in0, float* __restrict__ in1, float* __restrict__ out, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use grid-stride loop for better performance on large data
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < cnt; i += stride) {
        out[i] = in0[i] + in1[i];
    }
}