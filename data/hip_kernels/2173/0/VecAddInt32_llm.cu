#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAddInt32(int32_t* in0, int32_t* in1, int32_t* out, int cnt)
{
    // Use shared memory to reduce global memory accesses
    extern __shared__ int32_t shared_in0[];
    extern __shared__ int32_t shared_in1[];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    if (tid < cnt) {
        // Load data into shared memory
        shared_in0[local_tid] = in0[tid];
        shared_in1[local_tid] = in1[tid];
        __syncthreads(); // Synchronize to ensure data is loaded completely

        // Compute vector addition using shared memory
        out[tid] = shared_in0[local_tid] + shared_in1[local_tid];
    }
}