#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecSubInt32(int32_t* in0, int32_t* in1, int32_t* out, int cnt)
{
    // Calculate the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use loop to process multiple elements per thread
    for (int i = tid; i < cnt; i += blockDim.x * gridDim.x) {
        out[i] = in0[i] - in1[i];
    }
}