#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void prescan(float *g_odata, float *g_idata, int n)
{
    extern __shared__ float temp[];  // Shared memory
    int thid = threadIdx.x;
    int offset = 1;

    // Load input into shared memory
    int ai = 2 * thid;
    int bi = ai + 1;
    temp[ai] = g_idata[ai];
    temp[bi] = g_idata[bi];
    __syncthreads();  // Ensure all loads are done

    // Build sum in place up the tree
    for (int d = n >> 1; d > 0; d >>= 1)
    {
        if (thid < d)
        {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
        __syncthreads();  // Synchronize before next iteration
    }

    if (thid == 0) { temp[n - 1] = 0; } // Clear the last element

    // Traverse down tree & build scan
    for (int d = 1; d < n; d *= 2)
    {
        offset >>= 1;
        if (thid < d)
        {
            int ai = offset * (2 * thid + 1) - 1;
            int bi = offset * (2 * thid + 2) - 1;

            float t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
        __syncthreads();  // Synchronize before next iteration
    }
    
    // Write results to device memory
    g_odata[ai] = temp[ai];
    g_odata[bi] = temp[bi];
}