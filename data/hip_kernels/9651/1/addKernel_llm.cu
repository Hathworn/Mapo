#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel to handle more than one element per thread
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    int stride = gridDim.x * blockDim.x; // Calculate stride for grid handling
    for (; i < N; i += stride) // Loop over all elements with stride
    {
        c[i] = a[i] + b[i]; // Perform addition
    }
}