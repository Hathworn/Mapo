#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_mse ()
{
    __shared__ double partial_mse[REDUC_THREADS];
    int index = threadIdx.x;
    unsigned int n = d_ncases * d_ntarg;

    double sum_mse = 0.0;
    for (int i = blockIdx.x * blockDim.x + index; i < n; i += blockDim.x * gridDim.x) {
        double diff = d_output[i] - d_targets[i];
        sum_mse += diff * diff;
    }

    partial_mse[index] = sum_mse;
    __syncthreads();

    // Efficient reduction using unrolled loop
    for (int i = blockDim.x >> 1; i > 32; i >>= 1) {
        if (index < i) {
            partial_mse[index] += partial_mse[index + i];
        }
        __syncthreads();
    }

    // Unrolling the last warp
    if (index < 32) {
        volatile double* vsm = partial_mse;
        vsm[index] += vsm[index + 32];
        vsm[index] += vsm[index + 16];
        vsm[index] += vsm[index + 8];
        vsm[index] += vsm[index + 4];
        vsm[index] += vsm[index + 2];
        vsm[index] += vsm[index + 1];
    }

    if (index == 0) {
        d_mse_out[blockIdx.x] = partial_mse[0];
    }
}