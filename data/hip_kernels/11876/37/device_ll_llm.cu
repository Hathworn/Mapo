#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_ll() {
    __shared__ double partial_ll[REDUC_THREADS];
    int index = threadIdx.x;
    int n = d_ncases;
    int ntarg = d_ntarg;
    double sum_ll = 0.0;

    // Optimize loop stride calculation
    int stride = blockDim.x * gridDim.x;
    for (int i = blockIdx.x * blockDim.x + index; i < n; i += stride) {
        sum_ll -= log(d_output[i * ntarg + d_class[i]] + 1.e-30);
    }

    partial_ll[index] = sum_ll;
    __syncthreads();

    // Unroll reduction loop for improved performance
    for (int i = blockDim.x >> 1; i > 0; i >>= 1) {
        if (index < i) {
            partial_ll[index] += partial_ll[index + i];
        }
        __syncthreads();
    }

    if (index == 0) {
        d_mse_out[blockIdx.x] = partial_ll[0];
    }
}