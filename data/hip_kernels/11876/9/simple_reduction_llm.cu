#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_reduction(int *shared_var, int *input_values, int N, int iters)
{
    __shared__ int local_mem[256];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    // Directly use blockDim.x for local_dim calculation only when necessary
    int local_dim = (blockIdx.x * blockDim.x + blockDim.x > N) ? N - blockIdx.x * blockDim.x : blockDim.x;

    for (int iter = 0; iter < iters; iter++)
    {
        if (tid < N)
        {
            local_mem[local_tid] = input_values[tid];
        }

        __syncthreads(); // Ensure all threads have written to shared memory

        // Reduce in a tree-like manner for efficient parallel reduction
        for (int stride = 1; stride < local_dim; stride *= 2)
        {
            if (local_tid % (2 * stride) == 0 && local_tid + stride < local_dim)
            {
                local_mem[local_tid] += local_mem[local_tid + stride];
            }
            __syncthreads();
        }

        // Perform atomic add after reduction
        if (local_tid == 0)
        {
            atomicAdd(shared_var, local_mem[0]);
        }
        
        __syncthreads(); // Ensure correct completion of reduction before next iteration
    }
}