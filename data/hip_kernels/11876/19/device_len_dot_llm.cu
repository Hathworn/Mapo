#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_len_dot() {
    extern __shared__ float partial_sums[]; // Use dynamic shared memory
    float* partial_len = partial_sums;
    float* partial_dot = partial_sums + blockDim.x;

    int index = threadIdx.x;
    int n = d_n_inputs_cols * d_nhid;
    float sum_len = 0.0f;
    float sum_dot = 0.0f;

    for (int i = blockIdx.x * blockDim.x + index; i < n; i += blockDim.x * gridDim.x) {
        float w_grad = d_w_grad[i]; // Load d_w_grad once
        sum_len += w_grad * w_grad;
        float prev_grad = d_prev_grad[i]; // Load d_prev_grad once
        sum_dot += w_grad * prev_grad;
        d_prev_grad[i] = w_grad;
    }

    partial_len[index] = sum_len;
    partial_dot[index] = sum_dot;
    __syncthreads();

    // Use loop unrolling to reduce operations
    for (int i = blockDim.x >> 1; i > 0; i >>= 1) {
        if (index < i) {
            partial_len[index] += partial_len[index + i];
            partial_dot[index] += partial_dot[index + i];
        }
        __syncthreads();
    }

    if (index == 0) {
        d_len_out[blockIdx.x] = partial_len[0];
        d_dot_out[blockIdx.x] = partial_dot[0];
    }
}