#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_transpose() 
{
    int ivis = blockIdx.x * blockDim.x + threadIdx.x;
    int ihid = blockIdx.y;

    // Load data to shared memory to improve access coalescing
    __shared__ float blockTile[BLOCK_DIM][BLOCK_DIM+1];

    if (ivis < d_n_inputs)
        blockTile[threadIdx.y][threadIdx.x] = d_w[ihid*d_n_inputs_cols+ivis];

    __syncthreads();

    // Write transposed data to global memory
    if (threadIdx.y + blockIdx.y * blockDim.y < d_nhid_cols && ivis < d_n_inputs)
        d_wtr[ivis*d_nhid_cols+ihid] = blockTile[threadIdx.x][threadIdx.y];
}