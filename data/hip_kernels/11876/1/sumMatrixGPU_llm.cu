#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate unique thread index within grid
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix < nx && iy < ny) {
        // Compute linear index for matrices
        unsigned int idx = iy * nx + ix;
        // Perform element-wise addition
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}