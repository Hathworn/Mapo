#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate global thread IDs
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary check to ensure threads inside matrix boundaries
    if (ix < nx && iy < ny)
    {
        unsigned int idx = iy * nx + ix;  // Calculate linear index inside boundary check
        MatC[idx] = MatA[idx] + MatB[idx]; // Perform matrix addition
    }
}