#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Calculate row and column index for this thread
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread accesses valid indices
    if (i < num_rows && j < num_cols) {
        // Perform the square and assign
        float f = feats[i * ldf + j];
        feats_sq[i * lds + j] = f * f;
    }
}