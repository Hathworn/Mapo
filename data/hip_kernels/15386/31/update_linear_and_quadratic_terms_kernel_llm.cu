#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel( int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    float cur_weight = *cur_tot_weight;
    float new_num_frames = old_num_frames + cur_weight;
    float prior_scale_change = 1.0f;

    if(max_count != 0) {
        float old_prior_scale = max(old_num_frames, static_cast<float>(max_count)) / max_count;
        float new_prior_scale = max(new_num_frames, static_cast<float>(max_count)) / max_count;
        prior_scale_change += new_prior_scale - old_prior_scale;
    }

    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll the loop for better performance
    int32_t stride = blockDim.x * gridDim.x;
    for (; idx < n; idx += stride) {
        int32_t diag_idx = ((idx + 1) * (idx + 2) / 2) - 1;
        quadratic[diag_idx] += prior_scale_change;
    }

    // Use atomicAdd to prevent race conditions
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        atomicAdd(&linear[0], prior_offset * prior_scale_change);
    }
}