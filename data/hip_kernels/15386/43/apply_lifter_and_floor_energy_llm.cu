#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void apply_lifter_and_floor_energy( int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;
    
    float *feats = features + frame * ldf;

    // Apply lifter coefficients using shared memory to reduce global memory access
    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += blockDim.x) {
            float lift = lifter_coeffs[c];
            feats[c] *= lift; // Directly modify feats
        }
    }

    // Use thread 0 for each frame to apply energy
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = log(energy_floor);

        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        feats[0] = energy;
    }
}