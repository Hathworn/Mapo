#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float* A_in, int32_t ldi, float* A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float* Ar = A_in + block_id * ldi;
    float* Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    // Utilize shared memory for faster access in the loop
    __shared__ float2 shared[CU1DBLOCK];
    int idx = thread_id;

    while (idx < half_length) {
        // Ignore special case
        if (idx != 0) {
            // Load data into shared memory
            shared[thread_id] = reinterpret_cast<float2*>(Ar)[idx];
            __syncthreads();
            
            float2 val = shared[thread_id];
            float ret = val.x * val.x + val.y * val.y;
            Aw[idx] = use_power ? ret : sqrtf(ret);
        }
        idx += CU1DBLOCK;
    }

    // Handle special case
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}