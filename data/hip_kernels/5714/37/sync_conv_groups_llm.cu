#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Sync threads within a thread block
__global__ void sync_conv_groups() {
    __syncthreads(); // Use __syncthreads for intra-block synchronization
}