#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Use shared memory to optimize data access
    __shared__ float cache[256]; // Adjust size according to the blockSize
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;

    if (pos < len) {
        // Load data to shared memory
        cache[threadIdx.x] = d_src[pos];
        __syncthreads();

        // Use data from shared memory
        d_res[pos] = cache[threadIdx.x] * scale;
    }
}