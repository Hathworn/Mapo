#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    float cx = u[flow_row_offset + j] * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + static_cast<float>(i) + 1.0f;
    float px;
    float py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);
    float value = src[image_row_offset + j];
    float weight;

    // Use a loop to reduce repetition
    float corner_weights[4] = { dx * dy, (1.0f - dx) * dy, (1.0f - dx) * (1.0f - dy), dx * (1.0f - dy) };
    int x_offsets[4] = { 0, -1, -1, 0 };
    int y_offsets[4] = { 0, 0, -1, -1 };

    for (int idx = 0; idx < 4; ++idx) {
        int cur_tx = tx + x_offsets[idx];
        int cur_ty = ty + y_offsets[idx];
        if (cur_tx >= 0 && cur_tx < w && cur_ty >= 0 && cur_ty < h) {
            weight = corner_weights[idx];
            _atomicAdd(dst + cur_ty * image_stride + cur_tx, value * weight);
            _atomicAdd(normalization_factor + cur_ty * image_stride + cur_tx, weight);
        }
    }
}