#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate the 1D index for the thread
    int index = threadIdx.y * blockDim.x + threadIdx.x +
                (blockIdx.y * blockDim.y + blockIdx.x) * blockDim.x * blockDim.y;

    // Calculate total number of elements
    int totalElements = w * h;

    // Use 1D index to ensure threads work on unique positions
    if (index < totalElements)
    {
        image[index] = value;
    }
}