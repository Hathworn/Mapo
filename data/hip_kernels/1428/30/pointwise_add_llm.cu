#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pointwise_add(float *d_res, const float *d_op1, const float *d_op2, const int len)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilize shared memory for coalesced access
    __shared__ float s_op1[256];
    __shared__ float s_op2[256];

    if(pos < len) {
        // Load data into shared memory
        s_op1[threadIdx.x] = d_op1[pos];
        s_op2[threadIdx.x] = d_op2[pos];
        __syncthreads();

        // Perform pointwise addition
        d_res[pos] = s_op1[threadIdx.x] + s_op2[threadIdx.x];
    }
}