#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate thread index more efficiently
    int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (ptidx < npoints)
    {
        // Load data into registers for efficient access
        short2 loc = loc_[ptidx];

        // Perform scaled assignment directly
        x[ptidx] = __fmul_rn(loc.x, scale);
        y[ptidx] = __fmul_rn(loc.y, scale);
    }
}