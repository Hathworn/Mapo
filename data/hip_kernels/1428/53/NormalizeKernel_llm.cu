#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if indices are within bounds
    if (i < h && j < w) {
        const int pos = i * s + j;

        // Load scale once to register
        float scale = normalization_factor[pos];

        // Use conditional operator for inversion
        float invScale = (scale != 0.0f) ? (__frcp_rn(scale)) : 1.0f;

        // Apply normalization
        image[pos] *= invScale;
    }
}