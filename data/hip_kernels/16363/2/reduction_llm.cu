#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction(bool *B, int *number, int order) {
    int num = 0;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < order) {
        // Use a stride-based loop to enhance memory coalescing
        for (int i = idx; i < order * order; i += order) {
            if (B[i] == 1) {
                num++;
            }
        }
        number[idx] = num;
    }
}