#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logicalkernel(bool *A, bool *B, int *neighbours, int order, int degree) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < order) {
        // Loop invariants moved outside inner loop for optimization
        int baseIdxOrder = idx * order;
        for (int i = 0; i < degree; i++) {
            int n = neighbours[idx * degree + i];
            int nBaseIdxOrder = n * order;
            for (int j = 0; j < order; j++) {
                B[baseIdxOrder + j] = B[baseIdxOrder + j] || A[nBaseIdxOrder + j];
            }
        }
    }
}