#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() {
    // Utilize the built-in __syncthreads() function to synchronize threads within the block
    __syncthreads();
}