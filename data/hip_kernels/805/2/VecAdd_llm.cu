#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(int *a, int *b, int *c, int n) {
    // Using improved block and thread indexing
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Using loop unrolling for performance
    #pragma unroll
    for (; i < n; i += blockDim.x * gridDim.x) {
        c[i] = a[i] + b[i];
    }
}