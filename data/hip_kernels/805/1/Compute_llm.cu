#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Compute(int *a, int k, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate the id in the most optimized way
    int id = ((i / k) << 1) * k + k + (i % k); // Optimized calculation of id

    if (id < n) {
        a[id] += a[id - (id % k) - 1]; // Perform operation only if within bounds
    }
}