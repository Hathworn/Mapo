#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function
__global__ void decrement(char* current, char* answer) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Ensure global indexing
    if (i < N) { // Boundary check for the array to avoid out-of-bounds access
        answer[i] = current[i] - 1;  // Direct character arithmetic
    }
}