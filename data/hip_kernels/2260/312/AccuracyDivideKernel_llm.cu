#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory to buffer the `accuracy` value to avoid accessing global memory frequently
    __shared__ float shared_accuracy;

    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy;  // Load into shared memory by one thread
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        shared_accuracy /= N;  // Perform division in shared memory
        *accuracy = shared_accuracy;  // Write back the result by one thread
    }
}