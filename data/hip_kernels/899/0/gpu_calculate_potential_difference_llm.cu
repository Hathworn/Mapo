#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_calculate_potential_difference(const int size, const int number_of_rounds, const double* potential_per_atom, const double* potential_per_atom_temp, double* potential_difference)
{
    __shared__ double s_diff[1024];

    // Initialize shared memory
    s_diff[threadIdx.x] = 0.0;

    double diff = 0.0;

    // Use a single loop and reduce the number of accesses to global memory
    for (int n = threadIdx.x; n < size; n += blockDim.x) {
        if (n < size) {
            diff += potential_per_atom_temp[n] - potential_per_atom[n];
        }
    }

    // Store calculated difference in shared memory
    s_diff[threadIdx.x] = diff;
    __syncthreads();

    // Reduce sum within block using efficient reduction pattern
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            s_diff[threadIdx.x] += s_diff[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Write result from block to global memory
    if (threadIdx.x == 0) {
        atomicAdd(&potential_difference[0], s_diff[0]);
    }
}