#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_copy_velocity( const int num_atoms, const int offset, const int* __restrict__ g_group_contents, const double* __restrict__ g_vx_i, const double* __restrict__ g_vy_i, const double* __restrict__ g_vz_i, double* __restrict__ g_vx_o, double* __restrict__ g_vy_o, double* __restrict__ g_vz_o)
{
    const int n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n < num_atoms) {
        const int m = g_group_contents[offset + n];
        
        // Optimize memory access pattern by coalescing
        g_vx_o[n] = g_vx_i[m];
        g_vy_o[n] = g_vy_i[m];
        g_vz_o[n] = g_vz_i[m];
    }
}