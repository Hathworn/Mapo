#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_calculate_force_square_max(const int size, const int number_of_rounds, const double* __restrict__ force_per_atom, double* force_square_max)
{
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    __shared__ double s_force_square[1024];
    
    if (tid < 1024) {
        s_force_square[tid] = 0.0;
    }
    __syncthreads();
    
    double force_square = 0.0;

    for (int round = 0; round < number_of_rounds; ++round) {
        const int n = tid + round * 1024;
        if (n < size) {
            const double f = force_per_atom[n];
            force_square = max(force_square, f * f); // Use max function for performance
        }
    }

    if (tid < 1024) {
        s_force_square[tid] = force_square;
    }
    __syncthreads();

    // Optimized parallel reduction
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
        if (tid < offset) {
            s_force_square[tid] = max(s_force_square[tid], s_force_square[tid + offset]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        force_square_max[0] = s_force_square[0];
    }
}