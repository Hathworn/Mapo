#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_find_vac( const int num_atoms, const int correlation_step, const double* g_vx, const double* g_vy, const double* g_vz, const double* g_vx_all, const double* g_vy_all, const double* g_vz_all, double* g_vac_x, double* g_vac_y, double* g_vac_z)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int size_sum = bid * num_atoms;
    int number_of_rounds = (num_atoms - 1) / 128 + 1;
    __shared__ double s_vac_x[128];
    __shared__ double s_vac_y[128];
    __shared__ double s_vac_z[128];

    double vac_x = 0.0;
    double vac_y = 0.0;
    double vac_z = 0.0;

    // Loop unrolling for performance
    for (int round = 0; round < number_of_rounds; round += 2) {
        int n1 = tid + round * 128;
        int n2 = n1 + 128;
        
        if (n1 < num_atoms) {
            vac_x += g_vx[n1] * g_vx_all[size_sum + n1];
            vac_y += g_vy[n1] * g_vy_all[size_sum + n1];
            vac_z += g_vz[n1] * g_vz_all[size_sum + n1];
        }

        if (n2 < num_atoms) {
            vac_x += g_vx[n2] * g_vx_all[size_sum + n2];
            vac_y += g_vy[n2] * g_vy_all[size_sum + n2];
            vac_z += g_vz[n2] * g_vz_all[size_sum + n2];
        }
    }

    s_vac_x[tid] = vac_x;
    s_vac_y[tid] = vac_y;
    s_vac_z[tid] = vac_z;
    __syncthreads();

    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
        if (tid < offset) {
            s_vac_x[tid] += s_vac_x[tid + offset];
            s_vac_y[tid] += s_vac_y[tid + offset];
            s_vac_z[tid] += s_vac_z[tid + offset];
        }
        __syncthreads();
    }

    // Reduce memory accesses for updating global memory
    if (tid == 0) {
        int index = (bid <= correlation_step) ? (correlation_step - bid) : (correlation_step + gridDim.x - bid);
        atomicAdd(&g_vac_x[index], s_vac_x[0]);
        atomicAdd(&g_vac_y[index], s_vac_y[0]);
        atomicAdd(&g_vac_z[index], s_vac_z[0]);
    }
}