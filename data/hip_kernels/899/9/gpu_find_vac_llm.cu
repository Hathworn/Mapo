#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_find_vac(const int num_atoms, const int correlation_step, const int num_correlation_steps, const float* g_vx, const float* g_vy, const float* g_vz, const float* g_vx_all, const float* g_vy_all, const float* g_vz_all, float* g_vac_x, float* g_vac_y, float* g_vac_z) 
{
    const int num_atoms_sq = num_atoms * num_atoms;
    const int n1n2 = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early exit for out-of-bounds threads
    if (n1n2 >= num_atoms_sq)
        return;
    
    const int n1 = n1n2 / num_atoms;
    const int n2 = n1n2 - n1 * num_atoms;
    
    // Use temporary variables to reduce redundant calculations
    int base_index = n1 * num_atoms;
    
    for (int k = 0; k < num_correlation_steps; ++k) 
    {
        int nc = correlation_step - k;
        if (nc < 0)
            nc += num_correlation_steps;
        
        // Precompute array offsets
        int vac_offset = nc * num_atoms_sq + n1n2;
        int all_offset = k * num_atoms + n2;
        
        // Avoid redundant global memory accesses
        float vx_val = g_vx[n1];
        float vy_val = g_vy[n1];
        float vz_val = g_vz[n1];
        float vx_all_val = g_vx_all[all_offset];
        float vy_all_val = g_vy_all[all_offset];
        float vz_all_val = g_vz_all[all_offset];
        
        g_vac_x[vac_offset] += vx_val * vx_all_val;
        g_vac_y[vac_offset] += vy_val * vy_all_val;
        g_vac_z[vac_offset] += vz_val * vz_all_val;
    }
}