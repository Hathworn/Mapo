#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_copy_velocity(const int num_atoms, const double* __restrict__ g_vx_i, const double* __restrict__ g_vy_i, const double* __restrict__ g_vz_i, float* __restrict__ g_vx_o, float* __restrict__ g_vy_o, float* __restrict__ g_vz_o)
{
    const int n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n < num_atoms) {
        // Cast directly to float to avoid intermediate conversions
        g_vx_o[n] = static_cast<float>(g_vx_i[n]);
        g_vy_o[n] = static_cast<float>(g_vy_i[n]);
        g_vz_o[n] = static_cast<float>(g_vz_i[n]);
    }
}