#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_positions(const int size, const double position_step, const double* force_per_atom, const double* position_per_atom, double* position_per_atom_temp)
{
    const int n = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (n < size) {
        // Optimize by storing force in a register
        const double force = force_per_atom[n];
        
        // Combine operations to reduce memory access
        position_per_atom_temp[n] = position_per_atom[n] + force * position_step;
    }
}