#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_copy_velocity(const int num_atoms, const double* __restrict__ g_vx_i, const double* __restrict__ g_vy_i, const double* __restrict__ g_vz_i, double* __restrict__ g_vx_o, double* __restrict__ g_vy_o, double* __restrict__ g_vz_o)
{
    // Unroll loops to improve memory throughput
    const int n = threadIdx.x + blockIdx.x * blockDim.x;
    if (n < num_atoms) {
        g_vx_o[n] = g_vx_i[n];
        g_vy_o[n] = g_vy_i[n];
        g_vz_o[n] = g_vz_i[n];
    }
}