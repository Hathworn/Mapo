#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_copy_mass(const int num_atoms, const int* __restrict__ g_group_contents, const double* __restrict__ g_mass_i, double* __restrict__ g_mass_o)
{
    // Optimize: Use shared memory to minimize redundant memory access
    extern __shared__ int shared_group_contents[];

    const int tx = threadIdx.x;
    const int n = tx + blockIdx.x * blockDim.x;
    
    if (tx < num_atoms) {
        shared_group_contents[tx] = g_group_contents[tx + blockIdx.x * blockDim.x];
    }
    __syncthreads();
    
    if (n < num_atoms) {
        g_mass_o[n] = g_mass_i[shared_group_contents[tx]];
    }
}