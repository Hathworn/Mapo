#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_copy_velocity(const int num_atoms, const double* __restrict__ g_vx_i, const double* __restrict__ g_vy_i, const double* __restrict__ g_vz_i, double* __restrict__ g_vx_o, double* __restrict__ g_vy_o, double* __restrict__ g_vz_o)
{
    // Calculate global thread index
    const int n = threadIdx.x + blockIdx.x * blockDim.x;
    // Ensure thread is within bounds
    if (n < num_atoms) {
        // Copy velocities using memory coalescing
        g_vx_o[n] = g_vx_i[n];
        g_vy_o[n] = g_vy_i[n];
        g_vz_o[n] = g_vz_i[n];
    }
}