#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;
        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithoutReplacement(hiprandStateMtgp32* state, int totalSamples, int sample, float* dest, long distributions, int categories, float* origDist, float* normDistPrefixSum) {
    long curDistBase = blockIdx.x * blockDim.y + threadIdx.y;
    
    if (curDistBase < distributions) { // Limit work to available distributions
        // Each thread computes its own random number within the warp
        float r = hiprand_uniform(&state[blockIdx.x]);
        
        if (threadIdx.x == 0) { // Only first thread in warp performs the write and update
            int choice = binarySearchForMultinomial(
                normDistPrefixSum + curDistBase * categories,
                categories,
                r);

            dest[curDistBase * totalSamples + sample] = (float)choice + 1.0f; // Store result as 1-based index

            origDist[curDistBase * categories + choice] = 0.0f; // Set probability to 0 for no replacement
        }
    }
}