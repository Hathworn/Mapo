#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = (start + end) >> 1; // Optimized: Integer division replaced with bit shift
        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithReplacement(hiprandStateMtgp32* state, int totalSamples, float* dest, long distributions, int categories, float* normDistPrefixSum) {
    // Calculate current distribution index
    long curDist = blockIdx.x;

    while (curDist < distributions) {
        // Calculate sample index base for this block's work
        int sampleBase = (blockDim.y * blockIdx.y + threadIdx.y);

        while (sampleBase < totalSamples) {
            // Only compute if sample index is valid
            if (sampleBase < totalSamples) {
                // Each thread gets its own random number
                float r = hiprand_uniform(&state[blockIdx.x * blockDim.y + threadIdx.y]);

                // Threads in the warp execute the search independently for different indices
                int choice = binarySearchForMultinomial(normDistPrefixSum + curDist * categories, categories, r);

                // Increment the sample index directly
                int sample = sampleBase;
                // Write the result to the destination array
                dest[curDist * totalSamples + sample] = (float) choice + 1.0f;
            }

            // Move to the next set of samples handled by this block
            sampleBase += gridDim.y * blockDim.y;
        }

        // Move to the next distribution handled by this block
        curDist += gridDim.x;
    }
}