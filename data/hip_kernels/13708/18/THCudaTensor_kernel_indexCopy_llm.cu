#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexCopy(float *res, float *src, long* res_stride, float *index, long res_nDim, int dim, long idx_size, long src_size, long size_dim)
{
    // Calculate thread index with simplified formula
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    long flat_size = src_size / idx_size;

    if (thread_idx < flat_size)
    {
        for (int i = 0; i < idx_size; i++)
        {
            int leftover = thread_idx;
            int targetIdx = 0;
            int resIdx = 0;

            // Unroll innermost loop for performance optimization
            for (int d = 0; d < res_nDim; ++d)
            {
                if (d != dim)
                {
                    long stride_d = d < dim ? res_stride[d] / size_dim : res_stride[d];
                    long coeff = leftover / stride_d;
                    leftover -= coeff * stride_d;
                    long inc = coeff * res_stride[d];
                    targetIdx += inc; 
                    resIdx += inc;
                }
            }

            res[resIdx + ((int)(index[i]) - 1) * res_stride[dim]] = src[targetIdx + i * res_stride[dim]];
        }
    }
}