#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv2genericrev(float *input, float *kernel, float *output, int input_n, int input_h, int input_w, int kernel_n, int kernel_h, int kernel_w, float alpha, int stride_h, int stride_w)
{
    // output dimensions
    int output_h = input_h - (kernel_h - 1) * stride_h;
    int output_w = input_w - (kernel_w - 1) * stride_w;

    // this thread only processes one output, defined by the block Ids
    int kk = blockIdx.x;
    int ii = blockIdx.y;

    // batch id and kernel id
    int batch = blockIdx.z;
    int kid = threadIdx.x;

    // shared memory for output reduction
    extern __shared__ float shared_output[];

    // calculate output starting index for this thread
    int output_idx = (kk * input_n + batch) * output_h * output_w;

    // convolution loop
    for (int yy = threadIdx.y; yy < output_h; yy += blockDim.y) {
        for (int xx = kid; xx < output_w; xx += blockDim.x) {
            // Dot product in two dimensions (between input image and kernel)
            float sum = 0;
            for (int ky = 0; ky < kernel_h; ky++) {
                for (int kx = 0; kx < kernel_w; kx++) {
                    int input_x = xx * stride_w + kx;
                    int input_y = yy * stride_h + ky;
                    int input_idx = (ii * input_n + batch) * input_h * input_w + input_y * input_w + input_x;
                    int kernel_idx = (kk * kernel_n + ii) * kernel_h * kernel_w + ky * kernel_w + kx;
                    sum += input[input_idx] * kernel[kernel_idx];
                }
            }
            shared_output[yy * output_w + xx] = sum;
        }
    }

    __syncthreads();

    // reduce across block
    if (kid == 0) {
        for (int yy = 0; yy < output_h; yy++) {
            for (int xx = 0; xx < output_w; xx++) {
                float total = 0;
                for (int i = 0; i < blockDim.x; i++) {
                    total += shared_output[yy * output_w + i * output_w + xx];
                }
                output[output_idx + yy * output_w + xx] += alpha * total;
            }
        }
    }
}