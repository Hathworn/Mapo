#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexFill(float *tensor, long *stride, float *index, long src_nDim, int dim, long idx_size, long tensor_size, long size_dim, float val) 
{
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

    long flat_size = tensor_size / idx_size;

    if (thread_idx < flat_size) 
    {
        long leftover = thread_idx;
        long srcIdx = 0;

        // Compute source index based on flat index
        for (int d = 0; d < src_nDim; d++) 
        {
            long coeff;
            if (d < dim) 
            {
                coeff = leftover / (stride[d] / size_dim);
                leftover -= coeff * (stride[d] / size_dim);
            } 
            else if (d > dim) 
            {
                coeff = leftover / stride[d];
                leftover -= coeff * stride[d];
            }
            srcIdx += coeff * stride[d];
        }

        // Update tensor value for each index in idx_size
        for (int i = 0; i < idx_size; i++) 
        {
            long idx = static_cast<long>(index[i]) - 1;
            tensor[srcIdx + idx * stride[dim]] = val;
        }
    }
}