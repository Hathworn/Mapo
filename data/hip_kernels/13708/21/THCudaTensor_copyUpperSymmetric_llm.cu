#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_copyUpperSymmetric(float *input, int n, int len)
{
    // Calculate global index and total number of threads
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll loop for better memory coalescence and reduced iteration count
    for (; idx < len; idx += stride) {
        int r = idx % n;
        int c = idx / n;

        // Copy only if row index is greater than column index
        if (r > c) {
            input[idx] = input[r * n + c];
        }
    }
}