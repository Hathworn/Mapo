#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexSelect_contiguous(float *tensor, float *src, long stride, float *index, long idxSize) {
    // Launch a sufficient number of threads per block
    const int VT = 4;
    const int WARP_SIZE = 32;
    const int MAX_DIM_SIZE = 65535;
    const int blockStride = blockDim.y * MAX_DIM_SIZE;
    const int warpStride = VT * WARP_SIZE * MAX_DIM_SIZE;

    // Use grid-stride loops for better parallelization and scalability
    for (int idx = blockIdx.x * blockDim.y + threadIdx.y; idx < idxSize; idx += blockStride) {
        for (int startIdx = threadIdx.x + blockIdx.y * VT * WARP_SIZE; startIdx < stride; startIdx += warpStride) {
            const int srcIdx = ((int)index[idx] - 1) * stride;
            const int targetIdx = idx * stride;

            #pragma unroll
            for (int i = 0; i < VT; i++) {
                const int featureIdx = startIdx + i * WARP_SIZE;
                if (featureIdx < stride) {
                    tensor[targetIdx + featureIdx] = src[srcIdx + featureIdx];
                }
            }
        }
    }
}