#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;

typedef struct edge {
    int first, second;
} edges;

__global__ void bfs(const edge *edges, int *vertices, int current_depth) {
    int a = blockDim.x * blockIdx.x + threadIdx.x;
    if (a >= SIZ) return; // Ensure thread index doesn't exceed bounds

    int vfirst = edges[a].first;
    int dfirst = vertices[vfirst];
    int vsecond = edges[a].second;
    int dsecond = vertices[vsecond];

    // Coalesced memory accesses for vertices
    if ((dfirst == current_depth) && (dsecond == -1)) {
        vertices[vsecond] = dfirst + 1;
    }
    else if ((dfirst == -1) && (dsecond == current_depth)) {
        vertices[vfirst] = dsecond + 1;
    }
}