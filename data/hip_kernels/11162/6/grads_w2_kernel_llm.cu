#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;

typedef struct edge {
    int first, second;
} edges;

// Optimized kernel using 2D grid and block
__global__ void grads_w2_kernel(double * grads_W2, double * W2, double reg, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < size && j < size) { // Check bounds
        grads_W2[i * size + j] += W2[i * size + j] * reg;
    }
}