#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;

typedef struct edge {
    int first, second;
} edges;

__global__ void dscores_kernel_init(int * y, double * dscores, int size)
{
    // Optimize by calculating the linear index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Boundary check for valid index
    if (i < gridDim.x) {
        dscores[i * size + y[i]] -= 1;
    }
}