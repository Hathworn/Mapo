#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;

typedef struct edge {
    int first, second;
} edges;

__global__ void w2_kernel(double *grads_W2, double *W2, double learning_rate, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Compute unique index for elements
    if (idx < size * size) { // Ensure index is within bounds
        W2[idx] += -learning_rate * grads_W2[idx]; // Update W2
    }
}