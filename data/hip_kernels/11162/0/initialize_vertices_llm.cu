#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;

typedef struct edge {
    int first, second;
} edges;

__global__ void initialize_vertices(int *vertices, int starting_vertex) {
    int v = blockDim.x * blockIdx.x + threadIdx.x;
    if (v < SIZ) { // Ensure within bounds
        vertices[v] = (v == starting_vertex) ? 0 : -1; // Use ternary operator for compactness
    }
}