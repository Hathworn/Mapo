#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;

typedef struct edge {
    int first, second;
} edges;

__global__ void dhidden_cal_kernel(double * a1, double * dhidden, int size)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check for out-of-bounds access
    if (index < size)
    {
        // Perform the ReLU gradient operation
        dhidden[index] = a1[index] <= 0 ? 0 : dhidden[index];
    }
}