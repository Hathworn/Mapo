#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;

typedef struct edge {
    int first, second;
} edges;

__global__ void y_batch_kernel(double* y_batch, double* y, int* sample_indices, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < size) { // Ensure thread is within bounds
        y_batch[i] = y[sample_indices[i]];
    }
}