#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;

typedef struct edge {
    int first, second;
} edges;

__global__ void dscore_cal_kernel(double * dscores, int num_inputs, int size)
{
    // Optimized indexing for improved global memory coalescing
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_inputs * size) {
        dscores[idx] /= num_inputs;
    }
}