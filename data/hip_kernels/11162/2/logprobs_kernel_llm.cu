#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;

typedef struct edge {
    int first, second;
} edges;

__global__ void logprobs_kernel(double * corect_logprobs, double * probs, int* y, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Compute unique thread index
    if (i < size) { // Ensure index is within bounds
        corect_logprobs[i] = -log(probs[i * size + y[i]]); // Compute logprob
    }
}