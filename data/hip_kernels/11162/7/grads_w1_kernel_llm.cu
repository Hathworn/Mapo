#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;

typedef struct edge {
    int first, second;
} edges;

__global__ void grads_w1_kernel(double * grads_W1, double * W1, double reg, int size)
{
    // Compute global thread index
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (globalIdx < size * gridDim.x) // Ensure global index is within bounds
    {
        grads_W1[globalIdx] += W1[globalIdx] * reg; // Efficient computation using global thread index
    }
}