#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;

typedef struct edge {
    int first, second;
} edges;

__global__ void w1_kernel(double * grads_W1, double * W1, double learning_rate, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate 1D global thread index
    if (idx < size) { // Check bounds
        W1[idx] += (-learning_rate * grads_W1[idx]); // Update weights
    }
}