#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZ 20
#define num_inp 4

using namespace std;

typedef struct edge {
    int first, second;
} edges;

__global__ void x_batch_kernel(double* X_batch, double * X, int * sample_indices, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Compute global thread index
    if (i < size * gridDim.x) {                      // Check index within bounds
        int sample_idx = i / size;                   // Compute sample index
        int feature_idx = i % size;                  // Compute feature index
        X_batch[i] = X[sample_indices[sample_idx] * size + feature_idx];
    }
}