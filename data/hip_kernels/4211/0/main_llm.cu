#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "geo_invert_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    double2 *d_dot1 = NULL;
hipMalloc(&d_dot1, XSIZE*YSIZE);
double2 *d_dot2 = NULL;
hipMalloc(&d_dot2, XSIZE*YSIZE);
double *d_dist = NULL;
hipMalloc(&d_dist, XSIZE*YSIZE);
double *d_azimut = NULL;
hipMalloc(&d_azimut, XSIZE*YSIZE);
long count = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
geo_invert<<<gridBlock, threadBlock>>>(d_dot1,d_dot2,d_dist,d_azimut,count);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
geo_invert<<<gridBlock, threadBlock>>>(d_dot1,d_dot2,d_dist,d_azimut,count);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
geo_invert<<<gridBlock, threadBlock>>>(d_dot1,d_dot2,d_dist,d_azimut,count);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}