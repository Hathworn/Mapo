#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//#define ITEM_COUNT 2
#define _PI 3.14159265358979323846
#define _PI2 1.57079632679489661923
#define _RAD 6372795

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void geo_invert(double2* d_dot1, double2* d_dot2, double* d_dist, double* d_azimut, long count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (idx < count)
    {
        // Pre-compute radians conversion for dot1
        double lat1_radians = d_dot1[idx].x * _PI / 180;
        double lng1_radians = d_dot1[idx].y * _PI / 180;

        // Pre-compute radians conversion for dot2
        double lat2_radians = d_dot2[idx].x * _PI / 180;
        double lng2_radians = d_dot2[idx].y * _PI / 180;

        // Use pre-computed radians for trigonometric calculations
        double cl1 = cos(lat1_radians);
        double cl2 = cos(lat2_radians);
        double sl1 = sin(lat1_radians);
        double sl2 = sin(lat2_radians);
        double delta = lng2_radians - lng1_radians;
        double cdelta = cos(delta);
        double sdelta = sin(delta);

        // Calculate distance
        double y = sqrt(pow(cl2 * sdelta, 2) + pow(cl1 * sl2 - sl1 * cl2 * cdelta, 2));
        double x = sl1 * sl2 + cl1 * cl2 * cdelta;
        double ad = atan2(y, x); // Use atan2 for better precision in angle calculation
        d_dist[idx] = ad * _RAD;

        // Calculate azimuth
        x = (cl1 * sl2) - (sl1 * cl2 * cdelta);
        y = sdelta * cl2;

        double z;
        if (x == 0)
        {
            z = (y > 0) ? -90 : (y < 0) ? 90 : 0;
        }
        else
        {
            z = atan2(-y, x) * 180 / _PI; // Use atan2 for better handling of edge cases
            if (x < 0)
            {
                z += 180;
            }
        }

        double z2 = fmod(z + 180.0, 360.0) - 180.0; // Use fmod for angle wrapping

        z2 = -z2 * _PI / 180;
        double anglerad2 = fmod(z2, 2 * _PI); // Use fmod for wrapping
        d_azimut[idx] = anglerad2 * 180 / _PI;
    }
}