#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//#define ITEM_COUNT 2
#define _PI 3.14159265358979323846
#define _PI2 1.57079632679489661923
#define _RAD 6372795

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void d_cudainit(int *a, int *b)
{
    // Use blockIdx.x and blockDim.x for scalability
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unified condition check for all threads
    if (i < 2) 
    {
        b[i] = a[i] * 2;
    }
}