#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate unique thread index
    hiprand_init(seed, id, 0, &state[id]);  // Initialize each state uniquely
}