#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function using shared memory and loop unrolling
__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int stride = blockDim.x * gridDim.x;
    
    int64_t local_batch = batch_index;
    int64_t local_class = class_index;

    for (; idx < num_elements; idx += stride) {
        // Unroll the loop to enhance performance
        int64_t base_idx = idx * 3; // Precompute base index for efficiency
        to_normalize[base_idx] = local_batch;
        to_normalize[base_idx + 1] = local_class;
        to_normalize[base_idx + 2] = static_cast<int64_t>(original[idx]);
    }
}