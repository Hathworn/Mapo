#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory to reduce global memory transactions
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    __shared__ float shared_accuracy; // Declare shared memory
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy; // Load accuracy into shared memory
    }
    __syncthreads(); // Synchronize threads
    if (threadIdx.x == 0) {
        shared_accuracy /= N; // Perform division in shared memory
        *accuracy = shared_accuracy; // Store back the result to global memory
    }
}