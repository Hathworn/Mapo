#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FormPNGData(float3* colors, unsigned char* pixelData, unsigned int simWidth, unsigned int simHeight)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    int quadIdx = x + simWidth * y;

    // Thread blocks calculate positions outside boundaries
    if (x >= simWidth || y >= simHeight)
        return;

    // scale and map floating point pixel data [0.0, 1.0] to unsigned char*
    // data in [0, 255] for each RGB value
    unsigned char r = (unsigned char)(colors[4 * quadIdx].x * 255.0f);
    unsigned char g = (unsigned char)(colors[4 * quadIdx].y * 255.0f);
    unsigned char b = (unsigned char)(colors[4 * quadIdx].z * 255.0f);

    pixelData[3 * quadIdx + 0] = r;
    pixelData[3 * quadIdx + 1] = g;
    pixelData[3 * quadIdx + 2] = b;
}