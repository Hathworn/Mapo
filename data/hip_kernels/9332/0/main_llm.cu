#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "PopulateBins_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    int *rawBinSums = NULL;
hipMalloc(&rawBinSums, XSIZE*YSIZE);
float2 *rawRingPoints = NULL;
hipMalloc(&rawRingPoints, XSIZE*YSIZE);
float xRange = 1;
float yRange = 1;
int mode = 1;
int numPoints = 1;
unsigned int simHeight = 1;
unsigned int simWidth = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
PopulateBins<<<gridBlock, threadBlock>>>(rawBinSums,rawRingPoints,xRange,yRange,mode,numPoints,simHeight,simWidth);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
PopulateBins<<<gridBlock, threadBlock>>>(rawBinSums,rawRingPoints,xRange,yRange,mode,numPoints,simHeight,simWidth);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
PopulateBins<<<gridBlock, threadBlock>>>(rawBinSums,rawRingPoints,xRange,yRange,mode,numPoints,simHeight,simWidth);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}