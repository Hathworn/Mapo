#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PopulateBins(int* rawBinSums, float2* rawRingPoints, float xRange, float yRange, int mode, int numPoints, unsigned int simHeight, unsigned int simWidth)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= numPoints) return; // Ensure the thread idx is within bounds

    // Directly compute bin indices instead of using a loop
    int xBin = __float2int_rd((rawRingPoints[idx].x + xRange / 2.0f) * simWidth / xRange);
    int yBin = __float2int_rd((rawRingPoints[idx].y + yRange / 2.0f) * simHeight / yRange);

    // Clamp xBin and yBin to valid range
    xBin = max(0, min(simWidth - 1, xBin));
    yBin = max(0, min(simHeight - 1, yBin));

    if (idx == 10)
    {
        printf("Idx 10 reads X: %f, Y: %f\n", rawRingPoints[idx].x, rawRingPoints[idx].y);
        printf("Idx 10 trying to add to bin: %d, %d, %d\n", xBin, yBin, xBin + yBin * simWidth);
    }

    atomicAdd(&rawBinSums[xBin + yBin * simWidth], 1);
}