#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Zero_Histogram(int* rawBinSums, unsigned int simHeight, unsigned int simWidth)
{
    // Calculate global index
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    int quadIdx = x + y * simWidth;

    // Check bounds before accessing the array
    if (quadIdx < simHeight * simWidth)
    {
        rawBinSums[quadIdx] = 0; // Set the element to zero
    }
    else
    {
        printf("ERROR: Out of rawBinSums[] bounds, calling: %d, max: %d\n", quadIdx, simHeight * simWidth);
    }
}