#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "Color_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float3 *colors = NULL;
hipMalloc(&colors, XSIZE*YSIZE);
float3 *rawColorMap = NULL;
hipMalloc(&rawColorMap, XSIZE*YSIZE);
int *rawBinSums = NULL;
hipMalloc(&rawBinSums, XSIZE*YSIZE);
int mapMin = 1;
int mapMax = 1;
unsigned int simWidth = 1;
unsigned int simHeight = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
Color<<<gridBlock, threadBlock>>>(colors,rawColorMap,rawBinSums,mapMin,mapMax,simWidth,simHeight);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
Color<<<gridBlock, threadBlock>>>(colors,rawColorMap,rawBinSums,mapMin,mapMax,simWidth,simHeight);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
Color<<<gridBlock, threadBlock>>>(colors,rawColorMap,rawBinSums,mapMin,mapMax,simWidth,simHeight);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}