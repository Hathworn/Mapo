#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Color(float3* colors, float3* rawColorMap, int* rawBinSums, int mapMin, int mapMax, unsigned int simWidth, unsigned int simHeight)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    int quadIdx = x + simWidth * y;

    // Ensure quadIdx is within bounds
    if (quadIdx < simWidth * simHeight)
    {
        int mapped = (int)(0 + (((rawBinSums[quadIdx] - mapMin) * (511 - 0)) / (mapMax - mapMin)));

        // Use ternary operator for concise boundary checking and mapping
        float3 color = (mapped >= 511) ? rawColorMap[510] : 
                       (rawBinSums[quadIdx] <= 0) ? make_float3(0.15f, 0.15f, 0.16f) : 
                       rawColorMap[mapped];

        // Unroll the loop manually for better performance
        colors[4 * quadIdx] = color;
        colors[4 * quadIdx + 1] = color;
        colors[4 * quadIdx + 2] = color;
        colors[4 * quadIdx + 3] = color;
    }
}