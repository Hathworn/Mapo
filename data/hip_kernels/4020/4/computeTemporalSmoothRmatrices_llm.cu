#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeTemporalSmoothRmatrices(const float* Rmatrices, uint32_t numSamples, uint32_t subArraySize, uint32_t numSubArrays, const uint32_t* subArraySizes, uint32_t temporalSmoothing, float* TempRmatrices)
{
    // Use 1D blockIdx.y and threadIdx for better thread and memory access pattern
    int sampleIdx = blockIdx.x;
    int scanlineIdxLocal = blockIdx.y;
    int threadIndex = threadIdx.x + threadIdx.y * blockDim.x;
    int blockSize = blockDim.x * blockDim.y;

    if (sampleIdx < numSamples)
    {
        int subArraySizeLocal = subArraySizes[scanlineIdxLocal * numSamples + sampleIdx];
        if (subArraySizeLocal > 0)
        {
            int numelR = subArraySizeLocal * (subArraySizeLocal + 1) / 2;
            int numelRfull = subArraySize * (subArraySize + 1) / 2;

            // Calculate first and last index bounds for the temporal smoothing
            int firstIdx = max(0, sampleIdx - (int)temporalSmoothing) + scanlineIdxLocal * numSamples;
            int lastIdx = min((int)(numSamples) - 1, sampleIdx + (int)temporalSmoothing) + scanlineIdxLocal * numSamples;

            float scaling = 1.0f;

            for (int matrixIdx = threadIndex; matrixIdx < numelR; matrixIdx += blockSize)
            {
                float finalEntry = 0.0f;
                // Accumulate R matrices for temporal smoothing
                for (int tempIdx = firstIdx; tempIdx <= lastIdx; tempIdx++)
                {
                    finalEntry += Rmatrices[matrixIdx + tempIdx * numelRfull];
                }
                // Write the temporally smoothed result back
                TempRmatrices[matrixIdx + (scanlineIdxLocal * numSamples + sampleIdx) * numelRfull] = finalEntry * scaling;
            }
        }
    }
}