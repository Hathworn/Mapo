#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeTemporalSmoothRmatrices(const float* Rmatrices, uint32_t numSamples, uint32_t subArraySize, uint32_t numSubArrays, const uint32_t* subArraySizes, uint32_t temporalSmoothing, float* TempRmatrices)
{
    // Calculate thread index and sample index
    int tIdx = threadIdx.x + blockDim.x * threadIdx.y;
    int sampleIdx = blockIdx.x + blockIdx.y * gridDim.x;
    
    // Ensure we are within bounds
    if (sampleIdx >= numSamples) return;

    int subArraySizeLocal = subArraySizes[sampleIdx];
    int numelR = subArraySizeLocal * subArraySizeLocal;
    int numelRfull = subArraySize * subArraySize;

    // Calculate temporal index bounds
    int firstIdx = max(0, sampleIdx - (int)(temporalSmoothing));
    int lastIdx = min((int)(numSamples) - 1, sampleIdx + (int)(temporalSmoothing));

    // Scaling factor (keeping it but might be redundant)
    float scaling = 1.0f;

    // Optimize memory access by ensuring contiguous memory access through tIdx increment
    for (int matrixIdx = tIdx; matrixIdx < numelR; matrixIdx += blockDim.x * blockDim.y)
    {
        int colIdx = matrixIdx % subArraySizeLocal;
        int rowIdx = matrixIdx / subArraySizeLocal;
        int matrixStorageIdx = colIdx + rowIdx * subArraySize;

        float finalEntry = 0.0f;

        // Summing temporally smoothed R matrix values
        for (int tempIdx = firstIdx; tempIdx <= lastIdx; tempIdx++)
        {
            finalEntry += Rmatrices[matrixStorageIdx + tempIdx * numelRfull];
        }

        // Store the result
        TempRmatrices[matrixStorageIdx + sampleIdx * numelRfull] = finalEntry * scaling;
    }
}