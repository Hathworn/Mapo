#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fillUnusedDiagonal(float* Rmatrices, uint32_t numSamples, uint32_t subArraySize, const uint32_t* subArraySizes)
{
    // Calculate global thread index
    int tIdx = (blockIdx.y * gridDim.x + blockIdx.x) * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x + threadIdx.x);

    int sampleIdx = tIdx / (blockDim.x * blockDim.y);
    int localTIdx = tIdx % (blockDim.x * blockDim.y);

    if (sampleIdx < numSamples)
    {
        int subArraySizeLocal = subArraySizes[sampleIdx];
        int numelRfull = subArraySize * subArraySize;

        if (subArraySize > subArraySizeLocal)
        {
            float* R = &Rmatrices[sampleIdx * numelRfull];
            float diagEntry = R[subArraySize * subArraySize - 1];

            for (int diagIdx = subArraySizeLocal + localTIdx; diagIdx < subArraySize; diagIdx += blockDim.x * blockDim.y)
            {
                // Efficient calculation for diagonal index
                int matrixIdx = diagIdx * (subArraySize + 1);
                R[matrixIdx] = diagEntry;
            }
        }
    }
}