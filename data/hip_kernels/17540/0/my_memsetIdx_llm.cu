#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using shared memory
__global__ void my_memsetIdx(int* dg_array, int size, int scale) {
    extern __shared__ int shared_data[];

    const int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    const int tid = threadIdx.x;

    if (gtid < size) {
        shared_data[tid] = gtid * scale;  // Write to shared memory
        __syncthreads();  // Ensure all threads update shared memory
        dg_array[gtid] = shared_data[tid];  // Write from shared memory to global memory
    }
}