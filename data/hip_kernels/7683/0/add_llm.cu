#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void add(int a, int b, int *c) 
{
    // Use shared memory to reduce device memory access time
    extern __shared__ int sharedMem[];
    int threadId = threadIdx.x;

    // Store data in shared memory
    if (threadId == 0) {
        sharedMem[0] = a;
        sharedMem[1] = b;
    }

    // Synchronize threads to ensure shared memory is ready
    __syncthreads();

    // Each block calculates a sum independently
    if (threadId == 0) {
        *c = sharedMem[0] + sharedMem[1];
    }
}