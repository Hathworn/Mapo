#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_stencil(float *new_data, float *data, float *param_a, float *param_b, float *param_c, float *param_wrk, float *param_bnd) {

    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    // Early exit if index out of bounds
    if (_tid_ >= 129 * 65 * 65) return;

    int idx_0 = _tid_ / (65 * 65);
    int idx_1 = (_tid_ / 65) % 65;
    int idx_2 = _tid_ % 65;

    // Use bounds check to prevent redundant conditions
    if (idx_0 <= 0 || idx_0 >= 128 || idx_1 <= 0 || idx_1 >= 64 || idx_2 <= 0 || idx_2 >= 64) {
        new_data[_tid_] = 0.0; 
        return; 
    }

    int base_idx = idx_0 * 65 * 65 + idx_1 * 65 + idx_2; // Precompute base index

    // Fetch data values
    float v000 = data[base_idx];
    float v100 = data[base_idx + 65 * 65];
    float v010 = data[base_idx + 65];
    float v001 = data[base_idx + 1];
    float v110 = data[base_idx + 65 * 65 + 65];
    float v120 = data[base_idx + 65 * 65 - 65];
    float v210 = data[base_idx - 65 * 65 + 65];
    float v220 = data[base_idx - 65 * 65 - 65];
    float v011 = data[base_idx + 65 + 1];
    float v021 = data[base_idx - 65 + 1];
    float v012 = data[base_idx + 65 - 1];
    float v022 = data[base_idx - 65 - 1];
    float v101 = data[base_idx + 65 * 65 + 1];
    float v201 = data[base_idx - 65 * 65 + 1];
    float v102 = data[base_idx + 65 * 65 - 1];
    float v202 = data[base_idx - 65 * 65 - 1];
    float v200 = data[base_idx - 65 * 65];
    float v020 = data[base_idx - 65];
    float v002 = data[base_idx - 1];

    // Load parameters
    int param_a_idx = 65 * 65 * 4 * idx_0 + 65 * 4 * idx_1 + 4 * idx_2;
    int param_b_idx = 65 * 65 * 3 * idx_0 + 65 * 3 * idx_1 + 3 * idx_2;
    int param_c_idx = 65 * 65 * 3 * idx_0 + 65 * 3 * idx_1 + 3 * idx_2;
    int param_wrk_idx = 65 * 65 * idx_0 + 65 * idx_1 + idx_2;
    int param_bnd_idx = 65 * 65 * idx_0 + 65 * idx_1 + idx_2;

    // Compute new data value
    new_data[_tid_] =
        v000 + 0.8f * (((
        param_a[param_a_idx + 0] * v100 +
        param_a[param_a_idx + 1] * v010 +
        param_a[param_a_idx + 2] * v001 +
        param_b[param_b_idx + 0] * (v110 - v120 - v210 + v220) +
        param_b[param_b_idx + 1] * (v011 - v021 - v012 + v022) +
        param_b[param_b_idx + 2] * (v101 - v201 - v102 + v202) +
        param_c[param_c_idx + 0] * v200 +
        param_c[param_c_idx + 1] * v020 +
        param_c[param_c_idx + 2] * v002 +
        param_wrk[param_wrk_idx]) *
        param_a[param_a_idx + 3] - v000) * param_bnd[param_bnd_idx]);
}