#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_5(int *new_data, int *data)
{
    int _tid_ = threadIdx.x + blockIdx.x * blockDim.x;

    // Exit early if the thread index exceeds the array bounds
    if (_tid_ >= 10000000) return;

    // Precompute division result for efficiency, using bitwise operations for division
    int idx_2 = (_tid_ >> 1) % 500;

    // Perform the computation and write the result
    new_data[_tid_] = (data[_tid_] + idx_2) % 13377;
}