#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NUM_THREADS 511
#define ITERATIONS 100000

using namespace std;

__global__ void kernel_map(int *values, int *next_values)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a single check for boundary condition
    if (tid >= NUM_THREADS) return;
    
    next_values[tid] = values[tid] + 1;
}