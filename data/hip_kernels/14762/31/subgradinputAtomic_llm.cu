#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx_start = threadIdx.x;
    int xx_end = output_w;
    int xx_step = blockDim.x;

    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_end = output_h;
    int yy_step = blockDim.y * gridDim.y;

    // select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;

    // compute gradInput
    for(int yy = yy_start; yy < yy_end; yy += yy_step) {
        for(int xx = xx_start; xx < xx_end; xx += xx_step) {
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float z = gradOutput[yy * output_w + xx]; // Directly access the value
            float z_norm = z / float(kW * kH); // Precompute the normalized output

            for(int ky = 0; ky < kH; ky++) {
                for(int kx = 0; kx < kW; kx++) {
                    atomicAdd(&(ptr_gradInput[kx]), z_norm);
                }
                ptr_gradInput += input_w; // Move to the next row in input
            }
        }
    }
}