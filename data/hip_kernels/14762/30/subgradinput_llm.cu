#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinput(float *gradInput, float *gradOutput, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // iterators and shared memory
    int xx, yy, kx, ky;

    // output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int xx_start = threadIdx.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;

    // select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    
    // Loop coalescing optimization by swapping the inner for-loops with the outer ones
    for (yy = yy_start; yy < output_h; yy += blockDim.y * gridDim.y) {
        for (xx = xx_start; xx < output_w; xx += blockDim.x) {
            float z = gradOutput[yy * output_w + xx]; // Load gradOutput once

            for (ky = 0; ky < kH; ky++) {
                float *ptr_gradInput = gradInput + (yy * dH + ky) * input_w + xx * dW;

                for (kx = 0; kx < kW; kx++) {
                    // Accumulate result in gradInput
                    atomicAdd(&ptr_gradInput[kx], z / float(kW * kH));
                }
            }
        }
    }
}