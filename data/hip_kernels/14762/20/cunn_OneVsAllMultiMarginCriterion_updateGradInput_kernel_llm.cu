#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_OneVsAllMultiMarginCriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, int nframe, int dim, int sizeaverage, float *positiveWeight)
{
    int k = blockIdx.x;
    float *input_k = input + k*dim;
    float *gradInput_k = gradInput + k*dim;
    int target_k = ((int)target[k])-1;
    float g = (sizeaverage ? 1./((float)dim) : 1.);

    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;

    for (int i=i_start; i<i_end; i+=i_step)
    {
        float y = (i==target_k) ? 1.0 : -1.0;
        float z = 1 - input_k[i]*y;

        if(z > 0)
        {
            float weight = (i==target_k) ? positiveWeight[i] : 1.0;
            float h =  -y*g*weight;
            gradInput_k[i] = h;
        }
        else
            gradInput_k[i] = 0;
    }
    // No need for __syncthreads() or shared memory, as each thread handles separate segments
}