#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_OneVsAllNLLCriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, int nframe, int dim, int sizeaverage, float *positiveWeight)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= nframe * dim) return;  // Boundary check

    // Calculate frame index and dimension index
    int k = idx / dim;
    int i = idx % dim;

    float g = (sizeaverage ? 1. / ((float)nframe) : 1.);
    int target_k = ((int)target[k]) - 1;

    float *input_k = input + k * dim;
    float *gradInput_k = gradInput + k * dim;

    // Calculate gradient for current element
    float z = (i == target_k) ? -g * positiveWeight[i] / input_k[i] : g / (1 - input_k[i]);
    gradInput_k[i] = z;
}