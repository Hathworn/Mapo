#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_OneVsAllNLLCriterion_updateOutput_kernel(float *output, float *input, float *target, int nframe, int dim, int sizeaverage, float *positiveWeight)
{
    extern __shared__ float buffer[]; // Use dynamic shared memory for flexibility
    int k = blockIdx.x;
    float *input_k = input + k * dim;
    float *output_k = output + k;
    int target_k = ((int)target[k]) - 1;

    int idx = threadIdx.x;
    float sum = 0.0f;

    // Use loop unrolling for better performance
    for (int i = idx; i < dim; i += blockDim.x)
    {
        float z = (i == target_k) ? (-positiveWeight[i] * log(input_k[i])) : -logf(1 - input_k[i]);
        sum += z;
    }
    buffer[idx] = sum; // Store partial sum in shared memory
    __syncthreads();

    // Reduce sum using optimized parallel reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (idx < stride)
        {
            buffer[idx] += buffer[idx + stride];
        }
        __syncthreads();
    }

    // Apply the result to global memory
    if (idx == 0)
    {
        *output_k = sizeaverage ? buffer[0] / nframe : buffer[0];
    }
}