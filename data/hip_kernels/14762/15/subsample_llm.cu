#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = o % input_n;

    // Efficient calculation of global thread index
    int tx = threadIdx.x + blockDim.x * blockIdx.y;
    if (tx >= output_w * output_h) return;

    // Determine position in output grid
    int yy = tx / output_w;
    int xx = tx % output_w;

    // Select input/output plane
    float *ptr_output = output + o * output_w * output_h + yy * output_w + xx;
    float *ptr_input = input + i * input_w * input_h + yy * dH * input_w + xx * dW;

    // Get the good mask for (k, i) (k out, i in)
    float the_weight = weight[k];
    float the_bias = bias[k];

    // Compute the mean of the input image
    float sum = 0;
    for (int ky = 0; ky < kH; ky++) {
        for (int kx = 0; kx < kW; kx++) {
            sum += ptr_input[kx];
        }
        ptr_input += input_w; // Next input line
    }

    // Update output
    *ptr_output = the_weight * sum + the_bias;
}