#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinput(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW) {
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Identify output plane and weight
    int o = blockIdx.x; // output index
    int k = blockIdx.x % input_n;
    float the_weight = weight[k];

    // Adjust pointers for current thread's output and input
    gradOutput += o * output_w * output_h;
    gradInput += o * input_w * input_h;

    // Calculate starting positions and step sizes
    int xx_start = threadIdx.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int xx_step = blockDim.x;
    int yy_step = blockDim.y * gridDim.y;

    // Loop over output
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            // Compute input/output pointers
            float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;

            // Compute multiplication factor
            float z = *ptr_gradOutput * the_weight;

            // Accumulate over kernel area
            for (int ky = 0; ky < kH; ++ky) {
                for (int kx = 0; kx < kW; ++kx) {
                    ptr_gradInput[kx] += z;
                }
                ptr_gradInput += input_w;
            }
        }
    }
}