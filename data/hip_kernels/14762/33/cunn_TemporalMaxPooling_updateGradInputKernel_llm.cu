#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Reduce redundant calculations by computing offsets once
    int time_idx = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;
    if (time_idx < output_w) {
        int gradInput_offset = blockIdx.x * input_w * input_n + time_idx * input_n * dW;
        int gradOutput_offset = blockIdx.x * output_w * input_n + time_idx * input_n;
        int indices_offset = blockIdx.x * output_w * input_n + time_idx * input_n;

        // Load data into registers to reduce repeated global memory access
        float *gradOutput_data = gradOutput + gradOutput_offset;
        float *indices_data = indices + indices_offset;

        // Use loop unrolling for better performance
        #pragma unroll
        for (int feat = 0; feat < input_n; ++feat) {
            // Atomic addition to avoid race conditions
            atomicAdd(gradInput + gradInput_offset + (int)indices_data[feat] * input_n + feat, gradOutput_data[feat]);
        }
    }
}