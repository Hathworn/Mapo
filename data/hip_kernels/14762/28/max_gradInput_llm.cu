#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void max_gradInput(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Calculate the global thread index
    long o = blockIdx.x * blockDim.x + threadIdx.x;

    // Check to ensure the thread is within bounds
    if (o < nrows) {
        // Calculate the input offset
        long i = o * ncols;

        // Get the index from indices array, adjust for zero-based index
        long idx = static_cast<long>(indices[o]) - 1;

        // Perform the gradient update
        input[i + idx] = output[o];
    }
}