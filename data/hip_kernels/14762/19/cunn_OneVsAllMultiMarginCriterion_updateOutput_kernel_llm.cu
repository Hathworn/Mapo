#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_OneVsAllMultiMarginCriterion_updateOutput_kernel(float *output, float *input, float *target, int nframe, int dim, int sizeaverage, float *positiveWeight)
{
    __shared__ float buffer[MULTIMARGIN_THREADS];
    int k = blockIdx.x;
    float *input_k = input + k * dim;
    float *output_k = output + k;
    int target_k = ((int)target[k])-1;

    float threadSum = 0.0f;
    for(int i = threadIdx.x; i < dim; i += blockDim.x) // Use single loop for initialization and calculation
    {
        float y = (i == target_k) ? 1.0f : -1.0f; // Precompute y once per iteration
        float z = 1.0f - input_k[i] * y;
        if(z > 0.0f)
        {
            float weight = (i == target_k) ? positiveWeight[i] : 1.0f;
            threadSum += z * weight;
        }
    }
    buffer[threadIdx.x] = threadSum;
    __syncthreads();
    
    // Reduce with stride loop
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        if (threadIdx.x < stride)
        {
            buffer[threadIdx.x] += buffer[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        if(sizeaverage)
            *output_k = buffer[0] / dim;
        else
            *output_k = buffer[0];
    }
}