#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void max_output(float *input, float *output, float *indices, long nrows, long ncols)
{
    // output offset:
    long o = threadIdx.x + blockDim.x * blockIdx.x;
    if (o >= nrows) return;

    // input offset:
    long i = o * ncols;

    // compute max:
    float max = input[i];
    long argmax = 0;
    #pragma unroll
    for (long ii = 1; ii < ncols; ii++) {
        float val = input[i + ii];
        if (val > max) {
            max = val;
            argmax = ii;
        }
    }

    // store
    output[o] = max;
    indices[o] = argmax + 1;
}