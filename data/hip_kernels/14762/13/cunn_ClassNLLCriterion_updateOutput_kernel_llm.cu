#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_ClassNLLCriterion_updateOutput_kernel(float *output, float *input, float *target, int nframe, int ndim, int sizeAverage, int ntarget) {
    // Allocate shared memory for partial results
    __shared__ float shInputs[NTHREADS];
    
    // Current thread index
    int tid = threadIdx.x;
    
    // Initialize shared memory for this thread
    shInputs[tid] = 0.0f;

    // Loop over nframe with stride equal to block size to allow more threads to compute
    for (int i = tid; i < nframe; i += blockDim.x) {
        for (int j = 0; j < ntarget; ++j) {
            int t = static_cast<int>(target[i * ntarget + j]) - 1;  // Convert to zero-based index
            if (t >= 0) {
                shInputs[tid] += input[i * ndim + t];
            }
        }
    }

    // Synchronize to ensure all threads have computed their contributions
    __syncthreads();

    // Reduction step to sum all contributions in shared memory
    if (tid == 0) {
        // Initialize output
        float output_val = 0.0f;
        for (int i = 0; i < NTHREADS; ++i) {
            output_val += shInputs[i];
        }
        // Average if needed
        if (sizeAverage) {
            output_val /= nframe;
        }
        // Negate result as per original kernel logic
        *output = -output_val;
    }
}