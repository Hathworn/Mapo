#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate global thread indices
    int xx = blockIdx.x * blockDim.x + threadIdx.x;
    int yy = blockIdx.y * blockDim.y + threadIdx.y;

    // output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    if (xx < output_w && yy < output_h) { // Bounds checking
        // compute offsets based on blockIdx.x
        int o = blockIdx.x;
        int i = o;
        int k = blockIdx.x % input_n;

        // select input/output plane
        gradOutput = gradOutput + o * output_w * output_h;
        gradInput = gradInput + i * input_w * input_h;

        // get weight
        float the_weight = weight[k];

        // compute gradInput
        float *ptr_gradInput = gradInput + yy * dH * input_w + xx * dW;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float z = *ptr_gradOutput * the_weight;
        for (int ky = 0; ky < kH; ky++) {
            for (int kx = 0; kx < kW; kx++) {
                atomicAdd(&(ptr_gradInput[kx]), z); // Atomic add to avoid race conditions
            }
            ptr_gradInput += input_w; // Move to the next row
        }
    }
}