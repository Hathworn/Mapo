#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void atomicadaptivemaxgradinput( float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w )
{
    // Compute thread IDs
    int o = blockIdx.x;
    int i = o;
    int xx = threadIdx.x + blockIdx.y * blockDim.x;
    int yy = blockIdx.z * blockDim.y + threadIdx.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // Check if thread is within bounds
    if (xx < output_w && yy < output_h) {
        // Compute indices for input and output
        int y_start = static_cast<int>(floorf(float(yy) / output_h * input_h));
        int x_start = static_cast<int>(floorf(float(xx) / output_w * input_w));

        float *ptr_gradInput = gradInput + y_start * input_w + x_start;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float *ptr_ind_x = indices_x + yy * output_w + xx;
        float *ptr_ind_y = indices_y + yy * output_w + xx;
        float z = *ptr_gradOutput;

        int argmax_x = static_cast<int>(*ptr_ind_x) - 1;
        int argmax_y = static_cast<int>(*ptr_ind_y) - 1;

        // Atomic add since different threads could update the same variable
        atomicAdd(&ptr_gradInput[argmax_x + argmax_y * input_w], z);
    }
}