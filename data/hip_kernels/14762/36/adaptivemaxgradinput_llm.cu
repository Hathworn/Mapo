#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // Calculate offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;

    int xx = threadIdx.x;
    int yy = blockDim.y * blockIdx.y + threadIdx.y;
    
    extern __shared__ float shared_data[];

    // Select input/output plane
    gradOutput = gradOutput + o * output_w * output_h;
    gradInput = gradInput + i * input_w * input_h;
    indices_x = indices_x + o * output_w * output_h;
    indices_y = indices_y + o * output_w * output_h;

    // Compute gradInput
    while (yy < output_h) {
        int y_start = static_cast<int>(floorf(static_cast<float>(yy) / output_h * input_h));
        int x_start = static_cast<int>(floorf(static_cast<float>(xx) / output_w * input_w));
        
        float *ptr_gradInput = gradInput + y_start * input_w + x_start;
        float *ptr_gradOutput = gradOutput + yy * output_w + xx;
        float *ptr_ind_x = indices_x + yy * output_w + xx;
        float *ptr_ind_y = indices_y + yy * output_w + xx;
        
        // Pre-fetch gradOutput value to improve memory access pattern
        float z = __ldg(ptr_gradOutput);

        int argmax_x = __float2int_rd(*ptr_ind_x) - 1;
        int argmax_y = __float2int_rd(*ptr_ind_y) - 1;

        // Use atomic add to safely update from multiple threads
        atomicAdd(&ptr_gradInput[argmax_x + argmax_y * input_w], z);

        xx += blockDim.x;
        yy += blockDim.y * gridDim.y;
    }
}