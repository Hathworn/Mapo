#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_ClassNLLCriterion_updateGradInput_kernel(float *gradInput, float *target, int nframe, int ndim, float grad, int ntarget) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nframe) {  // Ensure we don't access out of bounds
        for (int j = 0; j < ntarget; ++j) {
            int t = static_cast<int>(target[i * ntarget + j]) - 1;
            if (t >= 0)  // Check for valid target index
                gradInput[i * ndim + t] = grad;
        }
    }
}