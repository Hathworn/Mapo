#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_LookupTable_accGradParametersKernel(float *input, float *indices, float *gradOutput, float *gradWeight, float *count, float defaultScale, long numel, long stride) {

    int idx = blockIdx.x * 4 + threadIdx.y;

    // Ensure the entire warp exits early if redundant input
    if (idx < numel && (idx == 0 || input[idx] != input[idx - 1])) {
        do {
            // Optimize shared memory access per warp
            const int startFeature = threadIdx.x + blockIdx.y * blockDim.x * 2;
            const int weightRow = ((int) input[idx] - 1) * stride;
            const int gradOutputRow = ((int) indices[idx] - 1) * stride;
            
            const float scale = count ? defaultScale / count[idx] : defaultScale;
            float gradient[2];
            float weight[2];

            #pragma unroll
            for (int ii = 0; ii < 2; ii++) { // Reduced unroll factor for better reg usage
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride) {
                    gradient[ii] = gradOutput[gradOutputRow + featureDim];
                    weight[ii] = gradWeight[weightRow + featureDim];
                }
            }

            #pragma unroll
            for (int ii = 0; ii < 2; ii++) { // Use the unroll for update
                weight[ii] += gradient[ii] * scale;
            }

            #pragma unroll
            for (int ii = 0; ii < 2; ii++) { // Optimize store operations
                int featureDim = startFeature + ii * WARP_SIZE;
                if (featureDim < stride) {
                    gradWeight[weightRow + featureDim] = weight[ii];
                }
            }

            idx++;
        } while (idx < numel && input[idx] == input[idx - 1]);
    }
}