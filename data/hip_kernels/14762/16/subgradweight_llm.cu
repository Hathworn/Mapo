#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradweight(float *input, float *gradOutput, float *gradWeight, float *gradBias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW, float scale)
{
    // Calculate output dimensions
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute thread block and thread ID
    int o = blockIdx.x;
    int k = blockIdx.x % input_n;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    // Calculate indices for starting positions and steps
    int yy_start = threadIdx.y, yy_end = output_h;
    int xx_start = threadIdx.x, xx_end = output_w;
    int xx_step = blockDim.x, yy_step = blockDim.y;

    // Point to the respective input and gradOutput planes
    gradOutput += o * output_w * output_h;
    input += o * input_w * input_h;

    // Shared memory for partial sums: gradWeight and gradBias computation
    __shared__ float shared_sums[2][CUDA_MAX_THREADS];
    shared_sums[0][tid] = 0.0f;  // For gradWeight
    shared_sums[1][tid] = 0.0f;  // For gradBias

    // Compute partial sums for gradWeight
    for (int yy = yy_start; yy < yy_end; yy += yy_step) {
        for (int xx = xx_start; xx < xx_end; xx += xx_step) {
            float *ptr_input = input + yy * dH * input_w + xx * dW;
            float z = gradOutput[yy * output_w + xx];
            for (int ky = 0; ky < kH; ++ky) {
                for (int kx = 0; kx < kW; ++kx) {
                    shared_sums[0][tid] += z * ptr_input[kx];
                }
                ptr_input += input_w;
            }
        }
    }
    __syncthreads();

    // Reduce for gradWeight
    if (tid == 0) {
        float total = 0;
        for (int i = 0; i < blockDim.x * blockDim.y; ++i) {
            total += shared_sums[0][i];
        }
        gradWeight[k] += scale * total;
    }
    __syncthreads();

    // Compute partial sums for gradBias
    for (int i = tid; i < output_w * output_h; i += blockDim.x * blockDim.y) {
        shared_sums[1][tid] += gradOutput[i];
    }
    __syncthreads();

    // Reduce for gradBias
    if (tid == 0) {
        float total = 0;
        for (int i = 0; i < blockDim.x * blockDim.y; ++i) {
            total += shared_sums[1][i];
        }
        gradBias[k] += scale * total;
    }
}