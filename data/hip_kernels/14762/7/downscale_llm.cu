#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    // Calculate global index of the thread
    long ii = blockIdx.x * blockDim.x + threadIdx.x + 
              (blockIdx.y * blockDim.y + threadIdx.y) * blockDim.x * gridDim.x;

    // Return if index is out of bounds
    if (ii >= no_elements) return;

    // Precompute scale factors
    int scaled_d2 = d2 * scale_factor;
    int scaled_d3 = d3 * scale_factor;

    // Iterate over scale factors and accumulate results
    for (int i = 0; i < scale_factor; i++) {
        for (int j = 0; j < scale_factor; j++) {
            int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
            gradInput_data[ii] += gradOutput_data[ipidx];
        }
    }
}
```
