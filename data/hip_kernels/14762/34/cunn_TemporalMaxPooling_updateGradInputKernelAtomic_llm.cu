#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate global thread index
    int globalThreadIdx = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;

    // Calculate the base offset
    int baseInputOffset = blockIdx.x * input_w * input_n + globalThreadIdx * input_n * dW;
    int baseOutputOffset = blockIdx.x * output_w * input_n + globalThreadIdx * input_n;
    
    if (globalThreadIdx < output_w) {
        // Access gradInput, gradOutput, and indices data directly using the base offsets
        float *gradInput_data = gradInput + baseInputOffset;
        float *gradOutput_data = gradOutput + baseOutputOffset;
        float *indices_data = indices + baseOutputOffset;
        
        // Optimize loop to increment pointers instead of using indexing in atomicAdd
        for (int feat = 0; feat < input_n; ++feat) {
            atomicAdd(&gradInput_data[(int)indices_data[feat] * input_n + feat], gradOutput_data[feat]);
        }
    }
}