#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_MSECriterion_updateOutput_kernel(float* output, float *input, float *target, int nframe, int dim, int sizeAverage)
{
    __shared__ float buffer[MSECRITERION_THREADS];
    int k = blockIdx.x;
    float *input_k = input + k*dim;
    float *target_k = target + k*dim;

    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;

    // Initialize buffer
    buffer[threadIdx.x] = 0.0f;

    // Compute the squared differences
    for (int i = i_start; i < i_end; i += i_step)
    {
        float z = input_k[i] - target_k[i];
        buffer[threadIdx.x] += z * z;
    }
    __syncthreads();

    // Reduction using first thread
    if (threadIdx.x == 0)
    {
        float sum = 0.0f;
        for (int i = 0; i < blockDim.x; i++)
        {
            sum += buffer[i];
        }
        // Option to average over dimensions
        if (sizeAverage)
            sum /= dim;

        // Assign to global memory output location of block
        atomicAdd(output, sum);
    }
}