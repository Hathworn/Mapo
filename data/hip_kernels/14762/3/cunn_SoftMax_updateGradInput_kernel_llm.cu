#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_SoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int nframe, int dim)
{
    __shared__ float buffer[SOFTMAX_THREADS];
    int k = blockIdx.x;
    float *gradInput_k = gradInput + k * dim;
    float *output_k = output + k * dim;
    float *gradOutput_k = gradOutput + k * dim;

    int i_start = threadIdx.x;
    int i_end = dim;
    int i_step = blockDim.x;

    // Initialize buffer
    float sum = 0.0f;
    for (int i = i_start; i < i_end; i += i_step) {
        sum += gradOutput_k[i] * output_k[i];
    }

    // Reduce sum across thread block
    buffer[threadIdx.x] = sum;
    __syncthreads();

    // Perform reduction using binary tree method
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            buffer[threadIdx.x] += buffer[threadIdx.x + stride];
        }
        __syncthreads();
    }

    sum = buffer[0];
    for (int i = i_start; i < i_end; i += i_step) {
        gradInput_k[i] = output_k[i] * (gradOutput_k[i] - sum);
    }
}