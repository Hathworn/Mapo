#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_MSECriterion_updateGradInput_kernel(float *gradInput, float *input, float *target, float norm, int nframe, int dim)
{
    int k = blockIdx.x;
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    // Calculate index for each thread in 1D
    if (i < nframe * dim) {
        gradInput[i] = norm * (input[i] - target[i]);
    }
}