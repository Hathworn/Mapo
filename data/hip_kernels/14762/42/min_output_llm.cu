#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void min_output(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Get global thread index
    long o = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Check if thread index is out of bounds
    if (o >= nrows) return;

    // Precompute input offset
    long i = o * ncols;

    // Initialize min and argmin with first element
    float min = input[i];
    long argmin = 0;

    // Efficient loop to find min and its index
    for (long ii = 1; ii < ncols; ii++) {
        float val = input[i + ii];
        if (val < min) {
            min = val;
            argmin = ii;
        }
    }

    // Store results
    output[o] = min;
    indices[o] = argmin + 1;
}