#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void min_gradInput(float *input, float *output, float *indices, long nrows, long ncols)
{
    // Calculate unique global index for the thread.
    long o = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure o is within bounds to avoid unnecessary calculations.
    if (o < nrows) {
        // Compute input offset.
        long i = o * ncols;

        // Retrieve index, adjust it, and update input.
        long idx = static_cast<long>(indices[o]) - 1;
        input[i + idx] = output[o];
    }
}