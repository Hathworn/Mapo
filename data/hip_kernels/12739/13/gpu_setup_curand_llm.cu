#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_setup_curand(uint64_t seed, hiprandState *curand_states, uint32_t num_engines) {
    uint64_t id = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure all threads get unique indices within the grid
    uint64_t stride = blockDim.x * gridDim.x;

    // Efficient iteration over engines using grid stride
    while(id < num_engines) {
        hiprand_init(id + seed, 0, 0, &curand_states[id]);
        id += stride;
    }
}