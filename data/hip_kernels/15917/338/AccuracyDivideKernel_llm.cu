#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load accuracy value into a local variable for faster access
    float local_accuracy = *accuracy;
    
    // Perform division
    local_accuracy /= N;
    
    // Update the global memory with the computed value once
    *accuracy = local_accuracy;
}