#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(int *a, int *b, int *c, int N)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within array bounds
    if (index < N) {
        c[index] = a[index] + b[index];
    }
}
```
