#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c)
{
    // Compute using each thread; assuming single block and thread for simplicity
    int idx = threadIdx.x + blockIdx.x * blockDim.x; 
    if (idx == 0) // Ensure only one thread writes the result
    {
        *c = a + b;
    }
}