#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (tid < N) { // Ensure thread index is within bounds
        c[tid] = a[tid] + b[tid];
    }
}