#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo) {
    // Use shared memory for histogram bins to reduce global memory accesses
    __shared__ unsigned int localHisto[256];
    
    // Initialize shared memory bins
    if (threadIdx.x < 256) {
        localHisto[threadIdx.x] = 0;
    }
    __syncthreads();

    // Calculate the starting index and the offset to the next block each thread will process
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (i < size) {
        atomicAdd(&localHisto[buffer[i]], 1);
        i += stride;
    }
    __syncthreads();

    // Write the shared memory results to global memory
    if (threadIdx.x < 256) {
        atomicAdd(&histo[threadIdx.x], localHisto[threadIdx.x]);
    }
}