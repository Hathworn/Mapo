#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop for better performance
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        c[i] = a[i] + b[i];
    }
}