#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_histogram(unsigned char *data, unsigned int *histogram)
{
    extern __shared__ unsigned int local_hist[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    // Initialize shared memory for local histogram
    local_hist[tid] = 0;
    __syncthreads(); // Ensure all threads have initialized

    // Populate local histogram
    while (i < N)
    {
        atomicAdd(&local_hist[data[i]], 1);
        i += blockDim.x * gridDim.x;
    }
    __syncthreads(); // Ensure local histogram is complete

    // Accumulate local histogram into global histogram
    atomicAdd(&histogram[tid], local_hist[tid]);
}