#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel( uchar4 *ptr, int ticks ) {
    // Use shared memory for intermediate calculations
    __shared__ float fx, fy, d;
    
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // Precompute commonly used constants
    float DIM_HALF = DIM / 2.0f; 

    // Calculate position-dependent variables
    fx = x - DIM_HALF;
    fy = y - DIM_HALF;
    d = sqrtf(fx * fx + fy * fy);
    
    // Compute grey value using precomputed variables
    float val = cosf(d / 10.0f - ticks / 7.0f) / (d / 10.0f + 1.0f);
    unsigned char grey = (unsigned char)(128.0f + 127.0f * val);
    
    // Assign values to the uchar4 structure
    ptr[offset].x = grey;
    ptr[offset].y = grey;
    ptr[offset].z = grey;
    ptr[offset].w = 255;
}