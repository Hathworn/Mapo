#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(int *a, int *b, int *temp, int *c)
{
    // Calculate global thread index
    int outputIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int result = 0;

    // Compute partial sum across this block
    for (int i = outputIndex; i < N; i += stride)
    {
        result += a[i] * b[i];
    }

    // Store result to shared memory
    temp[outputIndex] = result;

    // Synchronize threads within block
    __syncthreads();

    // Reduction within block
    for (int i = blockDim.x / 2; i > 0; i >>= 1)
    {
        if (threadIdx.x < i)
        {
            temp[outputIndex] += temp[outputIndex + i];
        }
        __syncthreads();
    }

    // Write block result to global memory
    if (threadIdx.x == 0)
    {
        c[blockIdx.x] = temp[outputIndex];
    }
}