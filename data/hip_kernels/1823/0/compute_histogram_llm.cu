#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 10000000

__global__ void compute_histogram(unsigned char *data, unsigned int *histogram)
{
    __shared__ unsigned int cache[256];
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    int i = blockIdx.x * blockDim.x + tid;

    // Initialize shared memory cache to zero
    if (tid < 256) {
        cache[tid] = 0;
    }
    __syncthreads();

    // Process data in a loop
    while (i < N) {
        atomicAdd(&cache[data[i]], 1);
        i += blockDim.x * gridDim.x;
    }
    __syncthreads();

    // Accumulate shared results into global histogram
    if (tid < 256) {
        atomicAdd(&histogram[tid], cache[tid]);
    }
}