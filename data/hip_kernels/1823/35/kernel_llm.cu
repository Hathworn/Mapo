#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(unsigned char *ptr, int ticks) {
    // Calculate thread's absolute position in 2D grid
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Exit if out of bounds
    if (x >= DIM || y >= DIM) return;

    // Calculate 1D offset index
    int offset = x + y * blockDim.x * gridDim.x;
    
    // Precompute common values for pixel calculation
    float fx = x - DIM/2;
    float fy = y - DIM/2;
    float d = sqrtf(fx * fx + fy * fy);
    float value = 128.0f + 127.0f * cosf(d/10.0f - ticks/7.0f) / (d/10.0f + 1.0f);
    unsigned char grey = (unsigned char)value;
    
    // Write result to memory with RGBA channels
    ptr[offset*4 + 0] = grey;
    ptr[offset*4 + 1] = grey;
    ptr[offset*4 + 2] = grey;
    ptr[offset*4 + 3] = 255;
}