#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add( int *a, int *b, int *c ) {
    // Optimize by using threadIdx.x for parallelism 
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = gridDim.x * blockDim.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += totalThreads;
    }
}