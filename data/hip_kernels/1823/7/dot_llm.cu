#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(int *a, int *b, int *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Prefetch memory globally to reduce global memory access
    for (; i < N; i += blockDim.x * gridDim.x) {
        c[i] = __ldg(&a[i]) * __ldg(&b[i]);
    }
}