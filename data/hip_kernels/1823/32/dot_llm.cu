#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    int stride = blockDim.x * gridDim.x;
    // Perform loop unrolling for better performance
    while (tid < N) {
        temp += a[tid] * b[tid];
        if (tid + stride < N) temp += a[tid + stride] * b[tid + stride];
        tid += 2 * stride; // double the stride for loop unrolling
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    // Perform reduction using a more efficient pattern
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (cacheIndex < offset) {
            cache[cacheIndex] += cache[cacheIndex + offset];
        }
        __syncthreads();
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}