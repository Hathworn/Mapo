#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot( int size, float *a, float *b, float *c ) {
    extern __shared__ float cache[]; // Use dynamic shared memory
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    
    float temp = 0;
    int stride = blockDim.x * gridDim.x; // Calculate stride once
    
    // Loop with improved stride access
    for (int idx = tid; idx < size; idx += stride) {
        temp += a[idx] * b[idx];
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    // Unroll the reduction loop
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}