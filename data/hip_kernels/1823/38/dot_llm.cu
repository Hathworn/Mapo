#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot(int size, float *a, float *b, float *c) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0;
    // Loop unrolling for better performance
    while (tid < size) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // Store temp result to shared memory
    cache[cacheIndex] = temp;

    // Synchronize threads in the block
    __syncthreads();

    // Shared memory reduction using two-way loop unrolling
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    // Store the block result in the global memory
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}