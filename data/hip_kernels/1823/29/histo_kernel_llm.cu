#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo) {
    // Shared memory for thread-local histogram
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();

    // Calculate global thread index and stride
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Accumulate local histogram
    while (i < size) {
        atomicAdd(&temp[buffer[i]], 1);
        i += stride;
    }
    __syncthreads();

    // Accumulate global histogram
    atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
}