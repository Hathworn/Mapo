#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum(int *a, int *b, int *c)
{
    // Using shared memory for faster access
    extern __shared__ int shared_mem[];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    // Load inputs into shared memory
    shared_mem[tid] = a[i] + b[i];
    __syncthreads();

    // Write result back to global memory
    c[i] = shared_mem[tid];
}