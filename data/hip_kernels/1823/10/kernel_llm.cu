#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        // Pre-compute mod values for reuse
        int idx1 = idx + 1;
        int idx2 = idx + 2;
        float as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) * 0.5f; // Use multiplication instead of division for efficiency
    }
}