#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot(int *a, int *b, int *c)
{
    // shared memory cache for partial sum results
    __shared__ int cache[THREADS_PER_BLOCK];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int result = 0;

    // unroll loop to improve performance, process two elements per iteration
    while(i < N)
    {
        result += a[i] * b[i];
        if (i + blockDim.x * gridDim.x < N)
            result += a[i + blockDim.x * gridDim.x] * b[i + blockDim.x * gridDim.x];
        i += 2 * blockDim.x * gridDim.x;
    }

    cache[threadIdx.x] = result;

    // wait for all other threads in the same block
    __syncthreads();

    // reduction step: optimized reduction using loop unrolling
    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if (threadIdx.x < offset)
        {
            cache[threadIdx.x] += cache[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // thread 0 writes the result for this block
    if (threadIdx.x == 0)
    {
        c[blockIdx.x] = cache[0];
    }
}