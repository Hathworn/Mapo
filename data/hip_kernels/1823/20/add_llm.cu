#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void add(int a, int b, int *c) {
    // Use thread 0 of block 0 to perform the addition to optimize for minimal launch overhead 
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *c = a + b;
    }
}