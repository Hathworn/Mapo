#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(int size, float *a, float *b, float *c) {
    extern __shared__ float cache[]; // Use extern shared memory for flexibility
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float temp = 0.0f;
    // Unroll the loop for better performance
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        temp += a[i] * b[i];
    }

    cache[cacheIndex] = temp;
    __syncthreads();

    // Optimize reduction using loop unrolling
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}