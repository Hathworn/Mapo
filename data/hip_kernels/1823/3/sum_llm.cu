#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(int *a, int *b, int *c, int N)
{
    // Calculate global index for threads
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds check to avoid illegal memory access
    if (i < N)
    {
        c[i] = a[i] + b[i];
    }
}