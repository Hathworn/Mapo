#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int addem(int a, int b) {
    return a + b;
}

__global__ void add(int a, int b, int *c) {
    // Synchronize to ensure all threads reach this point (not necessary for such small computation but good practice for more complex cases)
    __syncthreads();
    // Use thread number 0 to perform the operation for minimal warp divergence
    if (threadIdx.x == 0) {
        *c = addem(a, b);
    }
}