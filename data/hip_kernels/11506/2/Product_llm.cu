#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Product(float *a, float *b, float *c) {
    float cval = 0.00;
    int R = blockIdx.y * blockDim.y + threadIdx.y; // Row of the matrix
    int C = blockIdx.x * blockDim.x + threadIdx.x; // Column of the matrix
    int N = 1000; // Define the size of the matrix
    
    // Check if thread is within matrix bounds
    if (R < N && C < N) {
        // Use coalesced memory access for improved performance
        for (int j = 0; j < N; j++) {
            cval += a[R * N + j] * b[j * N + C];
        }
        c[R * N + C] = cval;
    }
}