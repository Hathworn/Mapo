#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Matrix_Product(double* A, double* g, double* C)
{
    // Use shared memory for 'g' to reduce global memory accesses
    extern __shared__ double g_shared[];
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int N = 1000;
    if (row >= N) return;
    
    double Cvalue = 0.0;
    int e = threadIdx.x;
    
    // Each block processes a portion of 'g'
    if (e < N) {
        g_shared[e] = g[e];
    }
    
    __syncthreads();
    
    // Compute the output value using shared memory
    for (int e = 0; e < N; e++) {
        Cvalue += A[N * row + e] * g_shared[e];
    }
    
    C[row] += Cvalue;
}