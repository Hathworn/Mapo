#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Check for out-of-bounds access
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}
```
