#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 1024  // Example size, adjust as needed

// Optimized global kernel function
__global__ void VecAdd(const float* A, const float* B, float* C, int numElements)
{
    // Calculate thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use threads efficiently by checking bounds
    if (i < numElements)
    {
        // Perform vector addition
        C[i] = A[i] + B[i];
    }
}