#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void testMemset(float* array, float value, int N) {
    // Calculate global index using block and thread indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if index is within bounds
    if (i < N) {
        array[i] = value;
    }
}