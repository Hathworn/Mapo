#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory to load data for efficient access
extern __shared__ float sharedAcc[];

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use block shared memory to reduce global memory access
    int tid = threadIdx.x;
    if (tid == 0) {
        sharedAcc[0] = *accuracy;
    }
    __syncthreads();

    // Have one thread perform the division
    if (tid == 0) {
        sharedAcc[0] /= N;
    }
    __syncthreads();

    // Write result back to global memory
    if (tid == 0) {
        *accuracy = sharedAcc[0];
    }
}