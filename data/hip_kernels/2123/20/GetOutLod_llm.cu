#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod, const size_t lod_len, size_t* out_lod0) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is within bounds before accessing arrays
    if (index < lod_len) {
        size_t in_idx = in_lod[index];
        // Access num_erased once and store in a register to reduce memory traffic
        size_t erased_value = num_erased[in_idx];
        out_lod0[index] = in_idx - erased_value;
    }
}