#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void innerProd(float *aa, float *bb, float *cc)
{
    __shared__ float temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure index is within bounds to avoid out-of-bounds access
    if (index < gridDim.x * blockDim.x) {
        temp[threadIdx.x] = aa[index] * bb[index];
    } else {
        temp[threadIdx.x] = 0.0f;
    }

    __syncthreads(); // Synchronize to ensure all threads have written to shared memory

    // Perform parallel reduction within the block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            temp[threadIdx.x] += temp[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Only the first thread of each block adds the result to cc
    if (threadIdx.x == 0) {
        atomicAdd(cc, temp[0]);
    }
}