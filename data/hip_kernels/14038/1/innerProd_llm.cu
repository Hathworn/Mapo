#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void innerProd(float *aa, float *bb, float *cc)
{
    __shared__ float temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Load data into shared memory
    float value = aa[index] * bb[index];
    temp[threadIdx.x] = value;
    
    __syncthreads();

    // Sum reduction within the block
    if (threadIdx.x == 0) {
        float sum = 0;
        for (int i = 0; i < THREADS_PER_BLOCK; i++) {
            sum += temp[i];
        }
        // Use atomicAdd to update global output safely
        atomicAdd(cc, sum);
    }
}