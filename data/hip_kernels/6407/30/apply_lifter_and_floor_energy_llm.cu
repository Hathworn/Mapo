#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define CU1DBLOCK 256

__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    __shared__ float shared_energy; // Shared memory for energy

    float *feats = features + frame * ldf;

    // Apply lifter coefficients using shared memory for improved memory access
    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += CU1DBLOCK) {
            if (c < num_cols) {
                float lift = lifter_coeffs[c];
                float f = feats[c];
                feats[c] = f * lift;
            }
        }
    }

    // Use energy; Thread 0 for each frame will apply energy
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = log(energy_floor);

        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        shared_energy = energy;
    }

    __syncthreads(); // Synchronize to ensure shared memory is updated

    if (use_energy && thread_id == 0) {
        feats[0] = shared_energy; // Sets the energy for the first feature
    }
}