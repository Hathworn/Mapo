#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    // Cache common indices and strides
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    int half_length = row_length / 2;
    int stride = blockDim.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    // Compute power spectrum for all indices in the stride
    for (int idx = thread_id; idx < half_length; idx += stride) {
        if (idx == 0) continue;  // Skip special case

        float2 val = reinterpret_cast<float2*>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }

    // Special case for first and last elements
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}