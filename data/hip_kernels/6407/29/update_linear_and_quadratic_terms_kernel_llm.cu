#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    float cur_weight = *cur_tot_weight;
    float new_num_frames = old_num_frames + cur_weight;
    float prior_scale_change = 1.0f;

    if (max_count != 0.0f) {
        float old_prior_scale = fmaxf(old_num_frames, (float)max_count) / max_count;
        float new_prior_scale = fmaxf(new_num_frames, (float)max_count) / max_count;
        prior_scale_change += new_prior_scale - old_prior_scale;
    }

    // Use shared memory to reduce global memory accesses
    __shared__ float partial_quadratic[256];  // Adjust size according to maximum blockDim.x
    int32_t tid = threadIdx.x;
    partial_quadratic[tid] = 0.0f;

    for (int32_t i = blockIdx.x * blockDim.x + tid; i < n; i += blockDim.x * gridDim.x) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        partial_quadratic[tid] += prior_scale_change;
    }
    __syncthreads();

    // Reduce results from partial_quadratic array
    if (tid == 0) {
        float sum = 0.0f;
        for (int32_t j = 0; j < blockDim.x; ++j) {
            sum += partial_quadratic[j];
        }
        atomicAdd(&quadratic[((blockIdx.x + 1) * (blockIdx.x + 2) / 2) - 1], sum);
    }

    // Update linear term only once
    if (tid == 0 && blockIdx.x == 0) {
        atomicAdd(&linear[0], prior_offset * prior_scale_change);
    }
}