#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use threadIdx.x to uniquely assign each thread to a task
    int tid = threadIdx.x;
    if (tid == 0) {  // Only a single thread needs to perform the division
        *accuracy /= N;
    }
}