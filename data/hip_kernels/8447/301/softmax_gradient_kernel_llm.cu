#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Unroll the loop for better performance
    for (int i = idx; i < dim; i += blockDim.x * 4) {
        tmp += (i < dim) ? dY[i] * Y[i] : 0.0f;
        tmp += (i + blockDim.x < dim) ? dY[i + blockDim.x] * Y[i + blockDim.x] : 0.0f;
        tmp += (i + 2 * blockDim.x < dim) ? dY[i + 2 * blockDim.x] * Y[i + 2 * blockDim.x] : 0.0f;
        tmp += (i + 3 * blockDim.x < dim) ? dY[i + 3 * blockDim.x] * Y[i + 3 * blockDim.x] : 0.0f;
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Use a more efficient reduction method
    for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
        if (idx < offset) {
            reduction_buffer[idx] += reduction_buffer[idx + offset];
        }
        __syncthreads();
    }

    // Compute gradient using final reduction result
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}