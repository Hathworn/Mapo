#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void double_value(double *x, double *y)
{
    // Use a flat index to handle larger arrays
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    y[idx] = 2.0 * x[idx];
}