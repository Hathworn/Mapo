#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    __shared__ float uni;
    // Load uniform values to shared memory using the first thread of each block
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];

    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x;

    __syncthreads();

    // Update g_data using conditional and single instruction instead of two adds
    if (threadIdx.x < n) {
        g_data[address] += uni;
        if (threadIdx.x + blockDim.x < n)
            g_data[address + blockDim.x] += uni;
    }
}