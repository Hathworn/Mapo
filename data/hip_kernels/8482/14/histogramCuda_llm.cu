#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogramCuda( const uint8_t * data, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * histogram )
{
    __shared__ uint32_t sharedHist[256]; // Declare shared memory for histogram bins
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (threadIdx.x < 256) { // Initialize shared histogram bins to 0
        sharedHist[threadIdx.x] = 0;
    }
    __syncthreads();

    if ( x < width && y < height ) {
        const uint32_t id = y * rowSize + x;
        atomicAdd(&sharedHist[data[id]], 1); // Update shared histogram
    }
    __syncthreads();

    if (threadIdx.x < 256) {
        atomicAdd(&histogram[threadIdx.x], sharedHist[threadIdx.x]); // Update global histogram
    }
}