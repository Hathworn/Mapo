#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseAndCuda(const uint8_t* in1, uint32_t rowSizeIn1, const uint8_t* in2, uint32_t rowSizeIn2, uint8_t* out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate the global thread index in x and y dimensions
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure the thread is within the boundaries of the image
    if (x < width && y < height) {
        const uint32_t idIn1 = y * rowSizeIn1 + x;
        const uint32_t idIn2 = y * rowSizeIn2 + x;
        const uint32_t idOut = y * rowSizeOut + x;

        // Perform bitwise AND operation
        out[idOut] = in1[idIn1] & in2[idIn2];
    }
}