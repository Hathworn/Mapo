#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void invertCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global thread ID
    const uint32_t index = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t stride = blockDim.x * gridDim.x;

    // Process each pixel in this thread's range
    for (uint32_t i = index; i < width * height; i += stride) {
        uint32_t x = i % width;
        uint32_t y = i / width;
        if (x < width && y < height) {
            out[y * rowSizeOut + x] = ~in[y * rowSizeIn + x];
        }
    }
}