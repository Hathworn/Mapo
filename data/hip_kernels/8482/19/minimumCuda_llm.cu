#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void minimumCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
    // Calculate global thread coordinates
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Check boundary condition
    if ( x < width && y < height ) {
        // Compute the linear index for each input and result
        const uint32_t idxIn1 = y * rowSizeIn1 + x;
        const uint32_t idxIn2 = y * rowSizeIn2 + x;
        const uint32_t idxOut = y * rowSizeOut + x;
        
        // Perform minimum operation and write output
        out[idxOut] = min(in1[idxIn1], in2[idxIn2]);
    }
}