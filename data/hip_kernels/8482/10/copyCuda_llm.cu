#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global row index using 1D block dimension to optimize parallel reading
    const uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate global column index using 1D block dimension to optimize parallel writing
    const uint32_t col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure that threads only attempt to access memory within bounds
    if (col < width && row < height) {
        out[row * rowSizeOut + col] = in[row * rowSizeIn + col];
    }
}