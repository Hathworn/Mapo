#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void maximumCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate unique index for each thread
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        // Calculate offset only once for efficiency
        uint32_t offset1 = y * rowSizeIn1 + x;
        uint32_t offset2 = y * rowSizeIn2 + x;
        uint32_t offsetOut = y * rowSizeOut + x;

        // Use offsets to access memory directly
        out[offsetOut] = (in1[offset1] > in2[offset2]) ? in1[offset1] : in2[offset2];
    }
}