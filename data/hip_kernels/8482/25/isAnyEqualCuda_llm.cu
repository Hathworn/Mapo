#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void isAnyEqualCuda(const uint8_t *image, uint8_t *value, size_t valueCount, uint32_t width, uint32_t height, uint32_t *differenceCount) 
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure only valid threads compute
    if (x < width && y < height) {
        // Calculate 1D index from 2D coordinates
        const uint32_t id = y * width + x;

        // Use shared memory for faster value access
        extern __shared__ uint8_t sharedValue[];
        if (threadIdx.x < valueCount) {
            sharedValue[threadIdx.x] = value[threadIdx.x];
        }
        __syncthreads();

        // Check for equality
        bool equal = false;
        for (uint32_t i = 0; i < valueCount; ++i) {
            if (image[id] == sharedValue[i]) {
                equal = true;
                break;
            }
        }

        // Use atomic operation for concurrent addition
        if (equal) 
            atomicAdd(differenceCount, 1);
    }
}