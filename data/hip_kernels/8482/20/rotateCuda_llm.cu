#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rotateCuda( const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, float inXStart, float inYStart, uint32_t width, uint32_t height, float cosAngle, float sinAngle )
{
    // Calculate output image coordinates
    uint32_t outX = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t outY = blockDim.y * blockIdx.y + threadIdx.y;

    // Only process valid output pixels
    if ( outX < width && outY < height ) {
        // Calculate exact input coordinates by inverse rotation
        const float exactInX = inXStart + cosAngle * outX + sinAngle * outY;
        const float exactInY = inYStart - sinAngle * outX + cosAngle * outY;

        const int32_t inX = static_cast<int32_t>(exactInX);
        const int32_t inY = static_cast<int32_t>(exactInY);

        // Calculate output pixel address
        uint8_t * outPixel = out + outY * rowSizeOut + outX;

        // Boundary check to ensure input pixels are within range
        if ( inX < 0 || inX >= width - 1 || inY < 0 || inY >= height - 1 ) {
            *outPixel = 0; // Set output pixel to 0 for out-of-bounds input
        }
        else {
            // Calculate input pixel address for bilinear interpolation
            const uint8_t * inPixel = in + inY * rowSizeIn + inX;

            // Perform bilinear interpolation
            const float probX = exactInX - inX;
            const float probY = exactInY - inY;
            const float mean = *inPixel * (1 - probX) * (1 - probY) +
                               *(inPixel + 1) * probX * (1 - probY) +
                               *(inPixel + rowSizeIn) * (1 - probX) * probY +
                               *(inPixel + rowSizeIn + 1) * probX * probY +
                               0.5f;

            // Assign computed pixel intensity to output
            *outPixel = static_cast<uint8_t>(mean);
        }
    }
}