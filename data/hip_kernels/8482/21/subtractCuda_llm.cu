#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtractCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
    // Calculate x and y indices from built-in variable
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if ( x < width && y < height ) {
        // Compute linear indices once
        uint32_t indexIn1 = y * rowSizeIn1 + x;
        uint32_t indexIn2 = y * rowSizeIn2 + x;
        uint32_t indexOut = y * rowSizeOut + x;

        // Perform subtraction with boundary condition
        out[indexOut] = (in1[indexIn1] > in2[indexIn2]) ? (in1[indexIn1] - in2[indexIn2]) : 0;
    }
}