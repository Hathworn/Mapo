#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void isEqualCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint32_t width, uint32_t height, uint32_t * isEqual )
{
    // Optimize index calculation by using 1D grid with 2D array
    const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t stride = blockDim.x * gridDim.x;
    
    for (uint32_t i = idx; i < width * height; i += stride) {
        const uint32_t x = i % width;
        const uint32_t y = i / width;
        const uint32_t partsEqual = static_cast<uint32_t>(in1[y * rowSizeIn1 + x] == in2[y * rowSizeIn2 + x]);
        atomicAnd(isEqual, partsEqual);
    }
}