#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void absoluteDifferenceCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height) {
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure we process only within bounds
    if (x < width && y < height) {
        // Use a single calculation and remove dereference to optimize
        uint32_t idx1 = y * rowSizeIn1 + x; // Compute once, use index
        uint32_t idx2 = y * rowSizeIn2 + x;
        uint32_t idxOut = y * rowSizeOut + x;
        out[idxOut] = abs(in1[idx1] - in2[idx2]); // Use built-in abs function for clarity
    }
}