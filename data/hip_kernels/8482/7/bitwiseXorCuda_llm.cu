#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseXorCuda(const uint8_t* __restrict__ in1, uint32_t rowSizeIn1, const uint8_t* __restrict__ in2, uint32_t rowSizeIn2, uint8_t* __restrict__ out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global thread ID
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if within bounds
    if (x < width && y < height) {
        // Calculate linear indices
        const uint32_t idIn1 = y * rowSizeIn1 + x;
        const uint32_t idIn2 = y * rowSizeIn2 + x;
        const uint32_t idOut = y * rowSizeOut + x;
        
        // Perform XOR
        out[idOut] = in1[idIn1] ^ in2[idIn2];
    }
}