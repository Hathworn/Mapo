#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseOrCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure x and y are within bounds
    if (x < width && y < height)
    {
        const uint32_t idIn1 = y * rowSizeIn1 + x;
        const uint32_t idIn2 = y * rowSizeIn2 + x;

        // Use initial value
        uint8_t val = in1[idIn1] | in2[idIn2];

        // Store value directly to output
        out[y * rowSizeOut + x] = val;
    }
}