#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lookupTableCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height, uint8_t *table)
{
    // Calculate global thread ID
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure the thread operates within the image boundaries
    if (x < width && y < height) {
        // Compute linear index for input and output arrays
        const uint32_t inIndex = y * rowSizeIn + x;
        const uint32_t outIndex = y * rowSizeOut + x;

        // Map input value through lookup table to output
        out[outIndex] = table[in[inIndex]];
    }
}