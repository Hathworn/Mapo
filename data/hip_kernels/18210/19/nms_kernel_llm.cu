#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float devIoU(const float *a, const float *b) {
    // a: [5, ] b: [5, ], ymin, xmin, ymax, xmax, score
    float w = max(0.0f, min(a[2], b[2]) - max(a[0], b[0]));
    float h = max(0.0f, min(a[3], b[3]) - max(a[1], b[1]));
    float intersect = w * h;
    float sa = (a[2] - a[0]) * (a[3] - a[1]);
    float sb = (b[2] - b[0]) * (b[3] - b[1]);
    float _union = sa + sb - intersect;
    float eps = 1e-4f;
    return intersect / (_union + eps);
}

__global__ void nms_kernel(float *bbox_dev, unsigned long long *mask_dev, int num_boxes, int col_blocks, float threshold) {
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;

    const int row_size = min(num_boxes - by * THREADS, THREADS);
    const int col_size = min(num_boxes - bx * THREADS, THREADS);

    __shared__ float sh[THREADS * 5];

    if (tx < col_size) {
        int cols = tx + bx * THREADS;
        #pragma unroll
        for (int j = 0; j < 5; j++) {
            sh[tx * 5 + j] = bbox_dev[cols * 5 + j];
        }
    }
    __syncthreads();

    if (tx < row_size) {
        const int cur_box_idx = (by * THREADS) + tx;
        float *cur_box = bbox_dev + cur_box_idx * 5;

        int start = (bx == by) ? tx + 1 : 0;
        unsigned long long t = 0;

        // Iterate only required times for reduced warp divergence
        for (int i = start; i < col_size; i++) {
            if (devIoU(cur_box, &sh[i * 5]) >= threshold) {
                t |= (1ULL << i);
            }
        }

        const int mask_idx = cur_box_idx * col_blocks + bx;
        mask_dev[mask_idx] = t;
    }
}
```
