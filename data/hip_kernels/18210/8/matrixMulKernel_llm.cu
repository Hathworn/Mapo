#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMulKernel(float *C, float *A, float *B, int width, int height) {
    // Define shared memory for tiles of A and B
    __shared__ float tileA[16][16];
    __shared__ float tileB[16][16];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = blockIdx.y * blockDim.y + ty;
    int col = blockIdx.x * blockDim.x + tx;

    float sum = 0;

    // Loop over all tiles
    for (int m = 0; m < (width - 1) / 16 + 1; ++m) {
        // Load data into shared memory if within bounds
        if (row < height && m * 16 + tx < width)
            tileA[ty][tx] = A[row * width + m * 16 + tx];
        else
            tileA[ty][tx] = 0;

        if (col < width && m * 16 + ty < height)
            tileB[ty][tx] = B[(m * 16 + ty) * width + col];
        else
            tileB[ty][tx] = 0;

        __syncthreads();

        // Compute using shared memory tiles
        for (int k = 0; k < 16; ++k)
            sum += tileA[ty][k] * tileB[k][tx];

        __syncthreads();
    }

    // Write result to global memory
    if (row < height && col < width)
        C[row * width + col] = sum;
}