#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate(float *da, float* ans_device, int N) {
    // Use shared memory for partial sum to reduce global memory access
    extern __shared__ float shared_da[];

    int tx = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tx;

    // Load data into shared memory
    if (idx < N) {
        shared_da[tx] = da[idx];
    } else {
        shared_da[tx] = 0.0f; // Handle boundary with zero padding
    }
    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tx < stride) {
            shared_da[tx] += shared_da[tx + stride];
        }
        __syncthreads();
    }

    // Write result from block 0 and thread 0 to global memory
    if (tx == 0 && blockIdx.x == 0) {
        ans_device[0] = shared_da[0];
    }
}