#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mat_transpose(const float *a, float *b, int n, int m){
    const int TILE_WIDTH = 8;
    __shared__ float temp[TILE_WIDTH][TILE_WIDTH+1]; // Use padding to avoid bank conflicts

    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int row = TILE_WIDTH * by + ty;
    int col = TILE_WIDTH * bx + tx;

    if (row < m && col < n) { // Boundary check to avoid accessing illegal memory
        temp[ty][tx] = a[row * n + col];
    }
    __syncthreads();

    row = TILE_WIDTH * bx + ty; // Transpose block offset
    col = TILE_WIDTH * by + tx;

    if (col < m && row < n) { // Boundary check for transposed indices
        b[col * n + row] = temp[tx][ty]; // Write transposed data
    }
}