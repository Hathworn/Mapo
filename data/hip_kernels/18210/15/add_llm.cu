#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel to handle more iterations per thread for reduced grid launching overhead
__global__ void add(const float *a, const float *b, float *c, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;  // Calculate stride to cover entire array

    // Iterate over all elements in the array with current thread's stride
    for (int i = idx; i < n; i += stride) {
        c[i] = a[i] + b[i];
    }
}