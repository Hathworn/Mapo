#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;
#define eps 1e-4

__global__ void convolution2d(float *img, float *kernel, float* result, int n, int m, int kw, int kh, int out_n, int out_m, bool padding)
{
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;
    int x = bx * blockDim.x + tx;
    int y = by * blockDim.y + ty;
    int idx = y * out_m + x;

    // Check boundary condition once
    if (x < out_m && y < out_n) {
        float ret = 0.0f;
        int half_kw = kw / 2, half_kh = kh / 2;  // Pre-calculate halves for efficiency
        int base_x = padding ? x - half_kw : x;
        int base_y = padding ? y - half_kh : y;

        // Use shared memory for kernel to reduce global memory access
        __shared__ float shared_kernel[32][32];  // Assuming a maximum kernel size of 32x32
        if (tx < kw && ty < kh) {
            shared_kernel[ty][tx] = kernel[ty * kw + tx];
        }
        __syncthreads();  // Synchronize to ensure all kernel elements are loaded

        // Iterate over the kernel dimensions
        for (int i = 0; i < kw; i++) {
            for (int j = 0; j < kh; j++) {
                int cur_x = base_x + i;
                int cur_y = base_y + j;
                if (cur_x >= 0 && cur_x < n && cur_y >= 0 && cur_y < m) {
                    ret += img[cur_y * m + cur_x] * shared_kernel[j][i];  // Use shared memory
                }
            }
        }

        result[idx] = ret;
    }
}