#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void showgrid() {
    // Use temporary variables to store thread and block indices
    int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z;
    int bx = blockIdx.x, by = blockIdx.y, bz = blockIdx.z;
    int bdx = blockDim.x, bdy = blockDim.y, bdz = blockDim.z;
    int gdx = gridDim.x, gdy = gridDim.y, gdz = gridDim.z;
    
    // Print using variables to optimize readability
    printf("thread: %d, %d %d\nblock Idxs: %d, %d %d\nblock Dims: %d, %d %d\ngrid: %d, %d %d\n\n\n",
           tx, ty, tz, bx, by, bz, bdx, bdy, bdz, gdx, gdy, gdz);
}