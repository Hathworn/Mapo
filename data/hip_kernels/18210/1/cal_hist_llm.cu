#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cal_hist(float *da, int *hist_da, int N, int M) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Check to ensure idx is within range
    if(idx < N) {
        // Use atomic operation for thread-safe addition
        atomicAdd(&hist_da[(int)da[idx]], 1);
    }
}