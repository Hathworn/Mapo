#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matadd(const float *a, const float *b, float *c, int n, int m) {
    // Calculate the global thread index in a 1D flattened grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * n;
    if (idx < n * m) {
        c[idx] = a[idx] + b[idx];
    }
}