#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matadd_1d(const float *a, const float *b, float *c, int n, int m) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < n) {
        // Unroll loop for better memory access and performance
        #pragma unroll
        for(int j = 0; j < m; j++) {
            int idx = j * n + i;
            c[idx] = a[idx] + b[idx];
        }
    }
}