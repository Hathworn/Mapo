#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmul_traditional(const float *a, const float *b, float *c, int n, int m) {
    // Calculate the global row and column index
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    // Proceed only if within bounds
    if(row < n && col < m) {
        float sum = 0.0f;
        // Use a single loop for clearer indexing
        for(int k = 0; k < n; ++k) {
            sum += a[row * n + k] * b[k * m + col];
        }
        c[row * m + col] = sum;
    }
}