#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmul_partition(const float *a, const float *b, float *c, int n) {
    const int TILE_WIDTH = 8;
    __shared__ float na[TILE_WIDTH][TILE_WIDTH];
    __shared__ float nb[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x, tx = threadIdx.x;
    int by = blockIdx.y, ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    float sum = 0.0f;

    // Loop over tiles
    #pragma unroll // Unroll the loop for better performance
    for (int m = 0; m < n / TILE_WIDTH; m++) {
        // Load the tile into shared memory
        na[ty][tx] = a[row * n + m * TILE_WIDTH + tx];
        nb[ty][tx] = b[(m * TILE_WIDTH + ty) * n + col];
        __syncthreads(); // Ensure all threads have loaded the tile

        #pragma unroll // Unroll the loop for better performance
        for (int k = 0; k < TILE_WIDTH; k++) {
            sum += na[ty][k] * nb[k][tx];
        }
        __syncthreads(); // Synchronize threads before loading new tile
    }

    // Write the computed result to the output matrix
    c[row * n + col] = sum;
}