#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmul(const float *a, const float *b, float *c, int n, int m) {
    // Assign thread indices for current computation
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Ensure thread is within matrix bounds
    if (i < n and j < m) {
        // Use shared memory for input matrices
        __shared__ float shared_a[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float shared_b[BLOCK_SIZE][BLOCK_SIZE];

        float sum = 0;

        // Loop over sub-matrices
        for (int tileIdx = 0; tileIdx < (n - 1) / BLOCK_SIZE + 1; ++tileIdx) {
            // Load data into shared memory
            if (j < m && tileIdx * BLOCK_SIZE + threadIdx.x < n)
                shared_a[threadIdx.y][threadIdx.x] = a[j * n + tileIdx * BLOCK_SIZE + threadIdx.x];
            if (i < n && tileIdx * BLOCK_SIZE + threadIdx.y < m)
                shared_b[threadIdx.y][threadIdx.x] = b[(tileIdx * BLOCK_SIZE + threadIdx.y) * n + i];
            
            __syncthreads(); // Synchronize threads to ensure data is loaded

            // Perform matrix multiplication on shared data
            for (int k = 0; k < BLOCK_SIZE; ++k) {
                sum += shared_a[threadIdx.y][k] * shared_b[k][threadIdx.x];
            }

            __syncthreads(); // Synchronize before loading new block
        }
        c[j * n + i] = sum; // Write result to global memory
    }
}