#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matadd_2d(const float *a, const float *b, float *c, int n, int m){
    // Calculate global thread index for 2D grid
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    // Check bounds for both dimensions
    if(i < n && j < m){
        int idx = j * n + i;
        // Perform addition
        c[idx] = a[idx] + b[idx];
    }
}