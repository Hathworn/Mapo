#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matadd(const float *a, const float *b, float *c, int n, int m) {
    // Calculate linear thread index in grid
    int idx = blockIdx.x * blockDim.x * m + blockIdx.y * blockDim.y + threadIdx.y * m + threadIdx.x; 
    int totalThreads = gridDim.x * blockDim.x * gridDim.y * blockDim.y;
    
    // Loop to process multiple elements by a single thread
    for (int i = idx; i < n * m; i += totalThreads) {
        c[i] = a[i] + b[i];
    }
}