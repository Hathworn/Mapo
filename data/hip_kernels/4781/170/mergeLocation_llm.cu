#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds
    if (ptidx >= npoints) return;

    short2 loc = loc_[ptidx];

    // Applying scale to x and y coordinates
    x[ptidx] = loc.x * scale;
    y[ptidx] = loc.y * scale;
}