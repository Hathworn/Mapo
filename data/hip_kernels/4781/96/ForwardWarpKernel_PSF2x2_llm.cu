#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];

    float weights[4];
    int txs[4] = {tx, tx-1, tx-1, tx};
    int tys[4] = {ty, ty, ty-1, ty-1};

    // Pre-calculate weights
    weights[0] = dx * dy;
    weights[1] = (1.0f - dx) * dy;
    weights[2] = (1.0f - dx) * (1.0f - dy);
    weights[3] = dx * (1.0f - dy);

    // Loop to handle atomic addition for four corners
    for (int k = 0; k < 4; k++)
    {
        if (txs[k] >= 0 && txs[k] < w && tys[k] >= 0 && tys[k] < h)
        {
            int idx = tys[k] * image_stride + txs[k];
            float weight = weights[k];
            _atomicAdd(dst + idx, value * weight);
            _atomicAdd(normalization_factor + idx, weight);
        }
    }
}