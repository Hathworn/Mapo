#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Use shared memory for better memory access patterns
    extern __shared__ float shared_scale[];

    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < h && j < w) {
        const int pos = i * s + j;
        // Load normalization factor to shared memory
        shared_scale[threadIdx.y * blockDim.x + threadIdx.x] = normalization_factor[pos];
        __syncthreads(); // Ensure all threads have loaded their parts

        float scale = shared_scale[threadIdx.y * blockDim.x + threadIdx.x];
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);
        image[pos] *= invScale;
    }
}