#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;
    for (int y = 0; y < h / 32; y++) {
        for (int x = 0; x < w; x += 16) {
            // Load minmax value into shared memory for efficient access
            __shared__ unsigned int shared_val[32];
            shared_val[tx] = minmax[y * w + x + tx];
            __syncthreads();

            unsigned int val = shared_val[tx];
            if (val) {
                // Unroll loop to reduce branch divergence and improve throughput
                #pragma unroll
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
            __syncthreads(); // Ensure all threads complete this block before continuing
        }
    }
}