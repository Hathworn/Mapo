#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputePositions(float *g_Data1, float *g_Data2, float *g_Data3, int *d_Ptrs, float *d_Sift, int numPts, int maxPts, int w, int h)
{
    int i = __mul24(blockIdx.x, POSBLK_SIZE) + threadIdx.x;
    if (i >= numPts) return;

    int p = d_Ptrs[i];
    float val[7];

    // Optimize memory access by reducing repeated indexing
    val[0] = g_Data2[p];
    float val2_p_minus_1 = g_Data2[p - 1];
    float val2_p_plus_1 = g_Data2[p + 1];
    float val2_p_minus_w = g_Data2[p - w];
    float val2_p_plus_w = g_Data2[p + w];

    // Use registers to store results
    float dx = 0.5f * (val2_p_plus_1 - val2_p_minus_1);
    float dxx = 2.0f * val[0] - val2_p_minus_1 - val2_p_plus_1;
    float dy = 0.5f * (val2_p_plus_w - val2_p_minus_w);
    float dyy = 2.0f * val[0] - val2_p_minus_w - val2_p_plus_w;

    val[5] = g_Data3[p];
    val[6] = g_Data1[p];
    
    float ds = 0.5f * (val[6] - val[5]);
    float dss = 2.0f * val[0] - val[5] - val[6];
    float dxy = 0.25f * (g_Data2[p + w + 1] + g_Data2[p - w - 1] - g_Data2[p - w + 1] - g_Data2[p + w - 1]);
    float dxs = 0.25f * (g_Data3[p + 1] + g_Data1[p - 1] - g_Data1[p + 1] - g_Data3[p - 1]);
    float dys = 0.25f * (g_Data3[p + w] + g_Data1[p - w] - g_Data3[p - w] - g_Data1[p + w]);

    // Calculate inverse determinant using precomputed values
    float idxx = dyy * dss - dys * dys;
    float idxy = dys * dxs - dxy * dss;
    float idxs = dxy * dys - dyy * dxs;
    float idyy = dxx * dss - dxs * dxs;
    float idys = dxy * dxs - dxx * dys;
    float idss = dxx * dyy - dxy * dxy;
    float det = idxx * dxx + idxy * dxy + idxs * dxs;
    float idet = 1.0f / det;

    // Use fast math for division
    float pdx = idet * (idxx * dx + idxy * dy + idxs * ds);
    float pdy = idet * (idxy * dx + idyy * dy + idys * ds);
    float pds = idet * (idxs * dx + idys * dy + idss * ds);

    if (pdx < -0.5f || pdx > 0.5f || pdy < -0.5f || pdy > 0.5f || pds < -0.5f || pds > 0.5f) {
        pdx = __fdividef(dx, dxx);
        pdy = __fdividef(dy, dyy);
        pds = __fdividef(ds, dss);
    }

    float dval = 0.5f * (dx * pdx + dy * pdy + ds * pds);
    d_Sift[i + 0 * maxPts] = (p % w) + pdx;
    d_Sift[i + 1 * maxPts] = (p / w) + pdy;
    d_Sift[i + 2 * maxPts] = d_ConstantA[0] * exp2f(pds * d_ConstantB[0]);
    d_Sift[i + 3 * maxPts] = val[0] + dval;
    
    float tra = dxx + dyy;
    det = dxx * dyy - dxy * dxy;
    d_Sift[i + 4 * maxPts] = __fdividef(tra * tra, det);
}