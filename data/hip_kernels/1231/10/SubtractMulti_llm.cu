#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    const int x = blockIdx.x * SUBTRACTM_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACTM_H + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;  // Calculate z index
    int sz = height * pitch;
    int p = z * sz + y * pitch + x;
    
    // Ensure indices are within bounds before accessing array
    if (x < width && y < height && z < gridDim.z)
    {
        // Perform subtraction operation
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
}