#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOrientations(float *g_Data, float *d_Sift, int maxPts, int w, int h) {
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    for (int i = 0; i < 13; i++)
        hist[i * 32 + tx] = 0.0f;
    __syncthreads();
    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    
    // Calculate Gaussian weights
    if (tx < 15)
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));
    
    int xp = (int)(d_Sift[bx + 0 * maxPts] - 6.5f);
    int yp = (int)(d_Sift[bx + 1 * maxPts] - 6.5f);
    int px = xp & 15;
    int x = tx - px;

    // Load image data into shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = min(max(0, xp + x), w - 1); // Clamping Xi within image bounds
        int yi = min(max(0, yp + y), h - 1); // Clamping Yi within image bounds
        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Calculate orientation histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = (int)(16.0f * atan2f(dy, dx) / 3.1416f + 16.5f) & 31;
            float grad = sqrtf(dx * dx + dy * dy);
            atomicAdd(&hist[32 * (x - 1) + bin], grad * gauss[x] * gauss[y]); // Use atomic add to avoid race condition
        }
    }
    __syncthreads();

    // Accumulate histogram values
    for (int stride = 8; stride > 0; stride >>= 1) {
        if (y < stride)
            hist[y * 32 + tx] += hist[(y + stride) * 32 + tx];
        __syncthreads();
    }

    // Refine histogram
    float v = hist[32 + tx];
    hist[tx] = (v > hist[32 + ((tx + 1) & 31)] && v >= hist[32 + ((tx + 31) & 31)] ? v : 0.0f);
    __syncthreads();

    if (tx == 0) {
        float maxval1 = 0.0, maxval2 = 0.0;
        int i1 = -1, i2 = -1;
        
        // Find the two strongest peaks
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }

        // Calculate the angle of the strongest peak
        float val1 = hist[32 + ((i1 + 1) & 31)];
        float val2 = hist[32 + ((i1 + 31) & 31)];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Sift[bx + 5 * maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);

        // Threshold for second peak
        if (maxval2 < 0.8f * maxval1)
            i2 = -1;

        // Calculate the angle of the second peak
        if (i2 >= 0) {
            val1 = hist[32 + ((i2 + 1) & 31)];
            val2 = hist[32 + ((i2 + 31) & 31)];
            peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Sift[bx + 6 * maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        } else {
            d_Sift[bx + 6 * maxPts] = -1.0f; // Indicate no valid second peak
        }
    }
}