#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void LowPassRowMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    __shared__ float data[CONVROW_W + 2*RADIUS];
    const int tx = threadIdx.x;
    const int block = blockIdx.x/(NUM_SCALES+3);
    const int scale = blockIdx.x - (NUM_SCALES+3)*block;
    const int xout = block*CONVROW_W + tx;
    const int loadPos = xout - RADIUS;
    const int yptr = blockIdx.y*pitch;
    const int writePos = yptr + height*pitch*scale + xout;
    float *kernel = d_Kernel + scale*16;

    // Use ternary operator for conditional loading
    data[tx] = (loadPos < 0) ? d_Data[yptr] : ((loadPos >= width) ? d_Data[yptr + width-1] : d_Data[yptr + loadPos]);

    __syncthreads();

    // Use loop unrolling for computation optimization
    if (xout < width && tx < CONVROW_W) {
        float sum = 0.0f;
        #pragma unroll
        for (int i = 0; i < 4; i++) {
            sum += (data[tx + i] + data[tx + (8 - i)]) * kernel[i];
        }
        sum += data[tx + 4] * kernel[4];
        d_Result[writePos] = sum;
    }
}