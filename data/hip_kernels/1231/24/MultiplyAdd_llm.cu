#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate the global index using blockDim and blockIdx directly
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Compute the linear index for the memory operation
    int p = y * width + x;
    
    // Check for boundary conditions
    if (x < width && y < height)
    {
        // Perform computation using constant memory and store the result
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
}