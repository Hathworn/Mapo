#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int pitch, int height)
{
    // Data cache
    __shared__ float data1[3 * (MINMAX_W + 2)];
    __shared__ float data2[3 * (MINMAX_W + 2)];
    __shared__ float data3[3 * (MINMAX_W + 2)];
    __shared__ float ymin1[(MINMAX_W + 2)];
    __shared__ float ymin2[(MINMAX_W + 2)];
    __shared__ float ymin3[(MINMAX_W + 2)];
    __shared__ float ymax1[(MINMAX_W + 2)];
    __shared__ float ymax2[(MINMAX_W + 2)];
    __shared__ float ymax3[(MINMAX_W + 2)];

    // Current tile and apron limits, relative to row start
    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_W;
    const int xEndClamped = min(xStart + MINMAX_W - 1, width - 1);
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;

    int memWid = MINMAX_W + 2;
    int memPos0 = tx - WARP_SIZE + 1;
    int memPos1 = memPos0;
    int yq = 0;
    unsigned int output = 0;
    
    for (int y = 0; y < 32 + 2; y++) {

        output >>= 1;
        int memPos = yq * memWid + memPos0;
        int yp = 32 * blockIdx.y + y - 1;
        yp = max(yp, 0);
        yp = min(yp, height - 1);
        int readStart = yp * pitch;

        // Load data into shared memory
        if (tx >= (WARP_SIZE - 1) && xReadPos >= 0 && xReadPos < width) {
            data1[memPos] = d_Data1[readStart + xReadPos];
            data2[memPos] = d_Data2[readStart + xReadPos];
            data3[memPos] = d_Data3[readStart + xReadPos];
        } else {
            data1[memPos] = data2[memPos] = data3[memPos] = 0;
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;
        if (y > 1 && tx < memWid) {
            // Calculate min and max
            float min1 = fminf(fminf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float min2 = fminf(fminf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float min3 = fminf(fminf(data3[memPos0], data3[memPos1]), data3[memPos2]);
            float max1 = fmaxf(fmaxf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float max2 = fmaxf(fmaxf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float max3 = fmaxf(fmaxf(data3[memPos0], data3[memPos1]), data3[memPos2]);
            ymin1[tx] = min1;
            ymin2[tx] = fminf(fminf(min1, min2), min3);
            ymin3[tx] = min3;
            ymax1[tx] = max1;
            ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
            ymax3[tx] = max3;
        }
        __syncthreads();

        if (y > 1 && tx < MINMAX_W && xWritePos <= xEndClamped) {
            // Calculate min and max considering thresholds
            float minv = fminf(fminf(fminf(fminf(fminf(ymin2[tx], ymin2[tx+2]), ymin1[tx+1]), ymin3[tx+1]), data2[memPos0+1]), data2[memPos2+1]);
            minv = fminf(minv, d_Threshold[1]);
            float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(ymax2[tx], ymax2[tx+2]), ymax1[tx+1]), ymax3[tx+1]), data2[memPos0+1]), data2[memPos2+1]);
            maxv = fmaxf(maxv, d_Threshold[0]);
            if (data2[memPos1+1] < minv || data2[memPos1+1] > maxv)
                output |= 0x80000000;
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }
    if (tx < MINMAX_W && xWritePos < width) {
        int writeStart = blockIdx.y * pitch + xWritePos;
        d_Result[writeStart] = output;
    }
}