#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    const int x = blockIdx.x * SUBTRACT_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACT_H + threadIdx.y;
    int p = y * pitch + x;

    // Remove __syncthreads() as it is not needed for this operation with independent threads
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}