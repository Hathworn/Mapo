#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;  // Simplified index calculation
    const int y = blockIdx.y * 16;
    int p = y * width + x;
    
    // Initialize shared arrays to the first value or a valid value from data
    float val = (x < width) ? d_Data[p] : d_Data[p - x];
    minvals[tx] = val;
    maxvals[tx] = val;

    // Loop to compute min/max values for each thread
    for (int ty = 1; ty < 16; ty++) {
        p += width;
        if (tx < width) {
            val = d_Data[p];
            minvals[tx] = min(minvals[tx], val);  // Use min() for clarity
            maxvals[tx] = max(maxvals[tx], val);  // Use max() for clarity
        }
    }
    __syncthreads();

    // Reduce operation to find block-level min and max
    for (int d = 1; d < blockDim.x; d <<= 1) {
        int mask = 2 * d - 1;  // Computation for striding
        if ((tx & mask) == 0 && (tx + d) < blockDim.x) {
            minvals[tx] = min(minvals[tx], minvals[tx + d]);
            maxvals[tx] = max(maxvals[tx], maxvals[tx + d]);
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}