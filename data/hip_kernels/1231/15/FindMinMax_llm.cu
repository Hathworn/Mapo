#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];

    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    const int b = blockDim.x;
    int p = y * pitch + x;

    float val = 0.0f;

    if (x < width) {
        val = d_Data[p];
    } else {
        val = d_Data[p - x]; // Avoid reading out of bounds
    }
    minvals[tx] = val;
    maxvals[tx] = val;

    // Unrolling loop for efficiency
    for (int ty = 0; ty < 15; ty++) { // Process 16 elements in steps
        p += pitch;
        if (x < width) {
            val = d_Data[p];
            if (val < minvals[tx])
                minvals[tx] = val;
            if (val > maxvals[tx])
                maxvals[tx] = val;
        }
    }
    
    __syncthreads();

    for (int d = b / 2; d > 0; d >>= 1) { // Reduce using shared memory
        if (tx < d) {
            if (minvals[tx + d] < minvals[tx])
                minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx])
                maxvals[tx] = maxvals[tx + d];
        }
        __syncthreads();
    }

    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr + 0] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}