#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;
    // Loop unrolling for outer y-loop
    #pragma unroll
    for (int y = 0; y < h / 32; y++) {
        for (int x = 0; x < w; x += 16) {
            unsigned int val = minmax[y * w + x + tx];
            if (val) {
                // Utilize bit Intrinsics for bit manipulation
                while (val && numPts < maxPts) {
                    int k = __ffs(val) - 1; // Find first set bit
                    if (k >= 0) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                        val &= val - 1; // Clear the least significant bit set
                    }
                }
            }
        }
    }
}