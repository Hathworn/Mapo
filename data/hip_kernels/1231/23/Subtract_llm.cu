#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Use more efficient way to calculate global thread index
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;

    // Check if the thread is within bounds
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}