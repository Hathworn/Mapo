#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Determine thread 0 in block 0 to perform division
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        *accuracy /= N;
    }
}