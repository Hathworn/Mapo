#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod, const size_t lod_len, size_t* out_lod0) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if index is within valid range before proceeding
    if (index < lod_len) {
        size_t lod_value = in_lod[index];
        // Avoid multiple indexing with `in_lod`
        out_lod0[index] = lod_value - num_erased[lod_value];
    }
}