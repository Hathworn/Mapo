#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to square elements of array 'a'
__global__ void square_array(float *a, int N)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds and unroll loop if feasible
    if (idx < N) {
        float val = a[idx];
        a[idx] = val * val;
    }
}