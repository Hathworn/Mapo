#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel for element solution assembly
__global__ void glob_sols(float *Le, float *w, float *u_glob, float *ue, int *cells, int num_cells)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = threadIdx.y;  // Use threadIdx.y directly
    int v;
    float Lii, weight;

    if(idx < num_cells && idy < 3) { // Limit idy within valid range (0-2)
        v = cells[idx * 3 + idy];  // getting global vertex number
        Lii = Le[idx * 9 + idy * 3 + idy];

        weight = Lii / w[v];

        atomicAdd(&u_glob[v], weight * ue[idx * 3 + idy]);
    }
}