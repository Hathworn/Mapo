#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void jacobi_iter(float *ue, float *up_glob, int *cells, float *temp1, int idx, int idy) {
    float ue_new;
    int v;
    int offset = 15 * threadIdx.x;

    v = cells[(idx * 3) + idy];
    
    // Register usage to minimize shared memory access
    float up_glob_v = up_glob[v];
    float temp1_base = temp1[offset + idy];

    ue_new = temp1[(offset + 9) + idy];
    temp1[(offset + 12) + idy] = up_glob_v;

    __syncthreads();

    // Use registers for repetitive access
    float t1_1 = temp1[offset + (idy * 3) + ((idy + 1) % 3)];
    float t1_2 = temp1[offset + (idy * 3) + ((idy + 2) % 3)];
    float t1_0 = temp1[offset + (idy * 3) + idy];

    ue_new -= t1_1 * temp1[(offset + 12) + (idy + 1) % 3];
    ue_new -= t1_2 * temp1[(offset + 12) + (idy + 2) % 3];

    // Avoid division if possible
    ue_new /= t1_0;

    ue[(idx * 3) + idy] = ue_new;
}

__device__ void elems_shared_cpy(float *Le, float *be, float *temp1, int idx, int idy) {
    int offset = 15 * threadIdx.x;

    // Prefetch shared memory data into registers if accessed more than once
    temp1[(offset + 9) + idy] = be[(idx * 3) + idy];
    for (int i = 0; i < 3; i++) {
        temp1[offset + (idy * 3) + i] = Le[(idx * 9) + (idy * 3) + i];
    }
}

__global__ void local_sols(float *Le, float *be, float *ue, float *up_glob, int *cells, int num_cells) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    extern __shared__ float temp1[];

    if (idx < num_cells && idy < blockDim.y) {
        elems_shared_cpy(Le, be, temp1, idx, idy);
        __syncthreads();
        jacobi_iter(ue, up_glob, cells, temp1, idx, idy);
    }
}