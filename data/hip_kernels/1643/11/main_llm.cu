#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "FluffyTail_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    const uint2 *source = NULL;
hipMalloc(&source, XSIZE*YSIZE);
uint2 *destination = NULL;
hipMalloc(&destination, XSIZE*YSIZE);
const int *sourceIndexes = NULL;
hipMalloc(&sourceIndexes, XSIZE*YSIZE);
int *destinationIndexes = NULL;
hipMalloc(&destinationIndexes, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
FluffyTail<<<gridBlock, threadBlock>>>(source,destination,sourceIndexes,destinationIndexes);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
FluffyTail<<<gridBlock, threadBlock>>>(source,destination,sourceIndexes,destinationIndexes);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
FluffyTail<<<gridBlock, threadBlock>>>(source,destination,sourceIndexes,destinationIndexes);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}