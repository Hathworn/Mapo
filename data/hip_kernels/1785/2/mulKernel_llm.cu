#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

enum ComputeMode { ADD, SUB, MUL, DIV };
hipError_t computeWithCuda(int *c, const int *a, const int *b, unsigned int size, ComputeMode mode);

__global__ void mulKernel(float* c, const float* a, const float* b)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure thread doesn't access out of bounds elements
    if (i < size) {
        c[i] = a[i] * b[i];
    }
}