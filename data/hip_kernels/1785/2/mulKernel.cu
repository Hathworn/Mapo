#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"



enum ComputeMode { ADD, SUB, MUL, DIV };
hipError_t computeWithCuda(int *c, const int *a, const int *b, unsigned int size, ComputeMode mode);

__global__ void mulKernel(float* c, const float* a, const float* b)
{
int i = threadIdx.x;
c[i] = a[i] * b[i];
}