#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

enum ComputeMode { ADD, SUB, MUL, DIV };
hipError_t computeWithCuda(int *c, const int *a, const int *b, unsigned int size, ComputeMode mode);

// Optimized kernel function
__global__ void dividKernel(float* c, const float* a, const float* b)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x; // Calculate global thread index
    if (i < size) // Boundary check to prevent out of range memory access
    {
        c[i] = a[i] / b[i];
    }
}