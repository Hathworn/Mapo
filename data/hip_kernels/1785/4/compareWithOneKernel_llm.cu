#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

enum ComputeMode { ADD, SUB, MUL, DIV };
hipError_t computeWithCuda(int *c, const int *a, const int *b, unsigned int size, ComputeMode mode);

__global__ void compareWithOneKernel(float* b, const double* a)
{
    // Calculate global thread index for processing larger arrays
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure that the current thread index is within bounds
    if (i < N) // Assuming N is the length of the arrays
    {
        if (a[i] == 1.0) // Use 1.0 for precision
        {
            b[i] += 1.0f; // Simplified increment operation
        }
    }
}