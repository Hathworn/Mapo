#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

enum ComputeMode { ADD, SUB, MUL, DIV };
hipError_t computeWithCuda(int *c, const int *a, const int *b, unsigned int size, ComputeMode mode);

// Optimized by using blockDim.x and blockIdx for scalable parallel processing.
__global__ void subKernel(float* c, const float* a, const float* b)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    c[i] = a[i] * b[i];
}