#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

enum ComputeMode { ADD, SUB, MUL, DIV };
hipError_t computeWithCuda(int *c, const int *a, const int *b, unsigned int size, ComputeMode mode);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform the computation if within bounds
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}