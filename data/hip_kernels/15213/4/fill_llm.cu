#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill(int * v, std::size_t size)
{
    // Cache commonly used values
    auto id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for better utilization
    for (std::size_t i = id; i < size; i += blockDim.x * gridDim.x)
    {
        v[i] = i;
    }
}