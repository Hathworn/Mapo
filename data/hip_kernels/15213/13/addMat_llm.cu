#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addMat(float* mA_d, float* mB_d, std::size_t w, std::size_t h)
{
    // Calculate single index for linear access pattern
    auto index = blockDim.x * blockIdx.x + threadIdx.x + (blockDim.y * blockIdx.y + threadIdx.y) * w;

    // Ensure index is within bounds for 1D operation optimization
    if (index < w * h)
    {
        mA_d[index] += mB_d[index];
    }
}