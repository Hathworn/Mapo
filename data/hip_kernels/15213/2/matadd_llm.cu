#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matadd(int *m0, int *m1, std::size_t w, std::size_t h) {
    // Calculate global thread id
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < w && j < h) {
        // Combine index computation for better readability and performance
        auto idx = i * w + j;
        m0[idx] += m1[idx];
    }
}