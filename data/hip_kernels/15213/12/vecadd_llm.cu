#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecadd(int * v0, int * v1, std::size_t size)
{
    // Calculate the global thread index
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop to process multiple elements per thread
    for (std::size_t i = tid; i < size; i += blockDim.x * gridDim.x)
    {
        v0[i] += v1[i];
    }
}