#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecadd(int *v0, int *v1, std::size_t size)
{
    // Calculate global thread index for larger vector sizes
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index does not exceed vector size
    if (tid < size)
    {
        v0[tid] += v1[tid];
    }
}