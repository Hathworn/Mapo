#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void grayscale(unsigned char *rgb, unsigned char *g, std::size_t cols, std::size_t rows) {
    // Calculate overall pixel index
    auto index = blockIdx.y * blockDim.y * cols + blockIdx.x * blockDim.x + threadIdx.y * cols + threadIdx.x;
    // Ensure index is within bounds
    if (index < cols * rows) {
        // Calculate grayscale value using predefined constants, avoid recomputation
        g[index] = (307 * rgb[3 * index] + 604 * rgb[3 * index + 1] + 113 * rgb[3 * index + 2]) / 1024;
    }
}