#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecadd(int* v0, int* v1, std::size_t size)
{
    auto tid = blockIdx.x * blockDim.x + threadIdx.x; // Use global thread index
    if (tid < size) // Check bounds to avoid out-of-range memory access
    {
        v0[tid] += v1[tid];
    }
}