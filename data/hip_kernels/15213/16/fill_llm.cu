#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill(int* v, std::size_t size)
{
    // Compute global thread ID
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure within bounds
    if (tid < size)
    {
        // Fill the array element
        v[tid] = tid;
    }
}