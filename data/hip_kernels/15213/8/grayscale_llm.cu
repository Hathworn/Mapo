#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void grayscale(unsigned char *rgb, unsigned char *g, std::size_t cols, std::size_t rows) {
    // Calculate global thread ID
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    auto j = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if thread is within image bounds
    if (i < cols && j < rows) {
        // Cache repeated calculation for efficiency
        auto index = 3 * (j * cols + i);
        // Update grayscale array
        g[j * cols + i] = (307 * rgb[index] + 604 * rgb[index + 1] + 113 * rgb[index + 2]) / 1024;
    }
}