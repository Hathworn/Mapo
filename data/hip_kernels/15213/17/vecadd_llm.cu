#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecadd(int *v0, int *v1, std::size_t size)
{
    // Calculate global thread index
    std::size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform operation only if within bounds
    if (tid < size) {
        v0[tid] += v1[tid];
    }
}