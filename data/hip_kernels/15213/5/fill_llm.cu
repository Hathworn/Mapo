#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill(int *v, std::size_t size)
{
    // Calculate the global thread ID
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Bounds check to prevent out-of-bounds access
    if (tid < size) {
        v[tid] = tid;
    }
}