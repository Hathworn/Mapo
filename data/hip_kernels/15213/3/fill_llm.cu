#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill(int *v, std::size_t size)
{
    auto tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID
    if (tid < size) // Boundary check
    {
        v[tid] = tid;
    }
}