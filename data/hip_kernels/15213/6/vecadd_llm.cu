#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecadd( int * v0, int * v1, std::size_t size )
{
    // Use blockIdx and blockDim to compute global index
    auto tid = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Ensure thread index is within bounds
    if (tid < size) 
    {
        v0[tid] += v1[tid];
    }
}