#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill(int * m, std::size_t w ,  std::size_t h)
{
    // Calculate global thread index
    auto idx = blockIdx.x * blockDim.x + threadIdx.x;
    auto idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check thread within matrix bounds and eliminate warp divergence by combining conditions
    if (idx < w && idy < h) {
        int linearIndex = idy * w + idx; // Calculate once to avoid redundant computation
        m[linearIndex] = linearIndex;    // Assign value directly
    }
}