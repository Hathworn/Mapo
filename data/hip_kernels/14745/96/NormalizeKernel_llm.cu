#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Compute global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within image bounds
    if (idx < w && idy < h) {
        int pos = idy * s + idx;

        // Load normalization factor and scale image
        float scale = normalization_factor[pos];
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

        image[pos] *= invScale;
    }
}