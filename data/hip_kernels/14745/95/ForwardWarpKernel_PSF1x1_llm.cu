#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Inline function to get the imaginary part of a float2
__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst) {
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    // Precompute offsets and read values from global memory
    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;
    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Compute the bottom left corner of the target pixel
    float cx = u_ * time_scale + (float)j + 1.0f;
    float cy = v_ * time_scale + (float)i + 1.0f;
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    float value = src[image_row_offset + j];
    
    // Atomic add only if target pixel is in bounds
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}