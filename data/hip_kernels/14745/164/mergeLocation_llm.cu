#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Load the location data once into local memory to reduce global memory access
        short2 loc = __ldg(&loc_[ptidx]);

        // Avoid re-computation by storing scaled values in registers
        float scaled_x = loc.x * scale;
        float scaled_y = loc.y * scale;
        
        // Store the results back to global memory
        x[ptidx] = scaled_x;
        y[ptidx] = scaled_y;
    }
}