#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit for out-of-bounds threads
    if (i >= h || j >= w) return;

    int pos = i * w + j;

    // Assign value to image position
    image[pos] = value;
}