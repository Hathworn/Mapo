#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Compute target pixel coordinates
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py, dx = modff(cx, &px), dy = modff(cy, &py);
    int tx = (int) px, ty = (int) py;
    float value = src[image_row_offset + j];

    // Precompute weights
    float weights[4] = { dx * dy, (1.0f - dx) * dy, (1.0f - dx) * (1.0f - dy), dx * (1.0f - dy) };
    int offsets[2][4] = { {0, -1, -1, 0}, {0, 0, -1, -1} };

    #pragma unroll 4 // Unroll the loop for efficiency
    for (int k = 0; k < 4; ++k) {
        int x = tx + offsets[0][k];
        int y = ty + offsets[1][k];
        if (x >= 0 && x < w && y >= 0 && y < h) {
            float weight = weights[k];
            _atomicAdd(dst + y * image_stride + x, value * weight);
            _atomicAdd(normalization_factor + y * image_stride + x, weight);
        }
    }
}