#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Iterate over elements using thread-stride loop
    for (int i = idx; i < size; i += stride) {
        r[i] = x[i] * y[i] + z[i];
    }
}