#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
const int scheme = 1; // 0 for explicit, 1 for implicit

__global__ void total(float *input, float *output, int len)
{
    // Load a segment of the input vector into shared memory
    __shared__ float partSum[BLOCK_SIZE];
    unsigned int tx = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * BLOCK_SIZE;
    
    float sum = 0; // Initialize sum for each thread

    // Load two elements per thread and reduce potential bank conflicts by using local sum
    if (start + tx < len) {
        sum += input[start + tx];
        if (start + BLOCK_SIZE + tx < len) sum += input[start + BLOCK_SIZE + tx];
    }

    partSum[tx] = sum;
    __syncthreads();

    // Traverse the reduction tree
    for (unsigned int stride = BLOCK_SIZE / 2; stride > 0; stride /= 2) {
        if (tx < stride) {
            partSum[tx] += partSum[tx + stride];
        }
        __syncthreads();
    }

    // Write the computed sum of the block to the output vector at the correct index
    if (tx == 0) {
        output[blockIdx.x] = partSum[0];
    }
}