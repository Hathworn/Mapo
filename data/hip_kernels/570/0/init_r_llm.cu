#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
const int scheme = 1; // 0 for explicit, 1 for implicit

/*************************************************************************************************************/
/*************								INITIALIZATION CODE										**********/
/*************************************************************************************************************/

__global__ void init_r(float* r, int N_cube) {
    // Calculate unique thread index within grid
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int plane_size = N_cube * N_cube; // Precompute plane size

    // Early return if out of bounds
    if (index >= N_ATOMS) return;

    // Calculate coordinates
    int iz = index / plane_size;
    int remainder = index % plane_size;
    int iy = remainder / N_cube;
    int ix = remainder % N_cube;

    // Compute positions based on grid indices
    r[index] = L / 2.0f * (1.0f - float(2 * ix + 1) / N_cube);
    r[index + N_ATOMS] = L / 2.0f * (1.0f - float(2 * iy + 1) / N_cube);
    r[index + 2 * N_ATOMS] = L / 2.0f * (1.0f - float(2 * iz + 1) / N_cube);
}