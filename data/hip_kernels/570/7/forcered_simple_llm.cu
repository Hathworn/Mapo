#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
// Include files

// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
const int scheme = 1; // 0 for explicit, 1 for implicit

/*************************************************************************************************************/
/*************								INITIALIZATION CODE										**********/
/*************************************************************************************************************/

__global__ void forcered_simple(float * force, float * forcered){
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    __shared__ float forcered_sh[3 * N_ATOMS];

    // Initialize shared memory with zero if within bounds.
    if (index < 3 * N_ATOMS) {
        forcered_sh[index] = 0.0f;
    }
    __syncthreads();

    // Calculate reduced force sums using atomic operations for thread safety.
    if (index < 3 * N_ATOMS) {
        int findex = (index / N_ATOMS) * N_ATOMS * N_ATOMS + (index % N_ATOMS);
        float sum = 0.0f;
        for (int i = 0; i < N_ATOMS; i++) {
            sum += force[findex + i * N_ATOMS];
        }
        atomicAdd(&forcered_sh[index], sum);
    }
    __syncthreads();

    // Write the results from shared to global memory if within bounds.
    if (index < 3 * N_ATOMS) {
        forcered[index] = forcered_sh[index];
    }
}