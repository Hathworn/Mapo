#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
const int scheme = 1; // 0 for explicit, 1 for implicit

/*************************************************************************************************************/
/*************								INITIALIZATION CODE										**********/
/*************************************************************************************************************/

__global__ void newForceReduction(float *input, float *output, int startunit, int len)
{
    unsigned int tx = threadIdx.x;
    unsigned int start = blockIdx.x * N_ATOMS;

    __shared__ float partSum[BLOCK_SIZE];

    // Initialize shared memory
    partSum[tx] = (tx < N_ATOMS) ? input[start + tx] : 0.0f;
    __syncthreads();

    // Unroll the loop for faster convergence
    // Use loop to eliminate repetitive code
    for (unsigned int stride = BLOCK_SIZE / 2; stride > 0; stride >>= 1) {
        if (tx < stride) {
            partSum[tx] += partSum[tx + stride];
        }
        __syncthreads();
    }

    // Store the result in global memory
    if (tx == 0) {
        output[blockIdx.x] = -partSum[0];
    }
}