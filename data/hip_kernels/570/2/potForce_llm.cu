#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Parameters

#define N_ATOMS 343
#define MASS_ATOM 1.0f
#define time_step 0.01f
#define L 10.5f
#define T 0.728f
#define NUM_STEPS 10000

const int BLOCK_SIZE = 1024;
const int scheme = 1; // 0 for explicit, 1 for implicit

__global__ void potForce(float * PairWise, int N, float * PotOut, float * ForceOut)
{
    // Cache common calculations to avoid redundant operations
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int row = tx + bx * BLOCK_SIZE;
    
    if (row < N * N)
    {
        float delx = PairWise[row];
        float dely = PairWise[row + N * N];
        float delz = PairWise[row + N * N * 2];
        float delr2 = delx * delx + dely * dely + delz * delz;

        if (delr2 == 0.0f) {
            PotOut[row] = 0.0f;
            ForceOut[row] = 0.0f;
            ForceOut[row + N * N] = 0.0f;
            ForceOut[row + N * N * 2] = 0.0f;
        } else {
            float delrm6 = __powf(delr2, -3.0f);
            float delrm12 = delrm6 * delrm6;
            float Pot = 4.0f * __fadd_rn(delrm12, -delrm6);
            float factor = 24.0f * __fadd_rn(2.0f * delrm12, -delrm6) / delr2;

            PotOut[row] = Pot;
            ForceOut[row] = -delx * factor;
            ForceOut[row + N * N] = -dely * factor;
            ForceOut[row + N * N * 2] = -delz * factor;
        }
    }
}