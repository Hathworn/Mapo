#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void freqencyStep1(char *d_dat, int len, int *d_freq) {
    // Load data into shared memory and reduce in parallel
    __shared__ int sfreq[3456]; // Shared memory for partial frequencies

    for (int i = threadIdx.x; i < 3456; i += blockDim.x) 
        sfreq[i] = 0; // Initialize shared memory
    __syncthreads();

    int *myfreq = &sfreq[27 * threadIdx.x];
    int stride = blockDim.x * gridDim.x;

    // Calculate frequency in shared memory
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < len; i += stride) 
        myfreq[d_dat[i] - 'a']++;
    __syncthreads();

    // Parallel reduction within shared memory
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            for (int i = 0; i < 26; i++)
                myfreq[i] += sfreq[27 * (threadIdx.x + offset) + i];
        }
        __syncthreads();
    }

    // Output results to global memory using atomics
    if (threadIdx.x < 26) 
        atomicAdd(&d_freq[threadIdx.x], myfreq[threadIdx.x]);
}