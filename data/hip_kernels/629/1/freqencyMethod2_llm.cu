#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void freqencyMethod2(char *d_dat, int len, int *d_freq) {
    __shared__ int sfreq[26];
    
    // Initialize shared memory using block-level cooperative approach
    for (int i = threadIdx.x; i < 26; i += blockDim.x) {
        sfreq[i] = 0;
    }
    __syncthreads();

    int gridsize = blockDim.x * gridDim.x;

    // Process elements with strided access for improved coalescing
    for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < len; i += gridsize) {
        int pos = d_dat[i] - 'a';
        atomicAdd(&sfreq[pos], 1);
    }
    __syncthreads();

    // Accumulate results from shared memory to global memory
    for (int i = threadIdx.x; i < 26; i += blockDim.x) {
        atomicAdd(&d_freq[i], sfreq[i]);
    }
}