#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSoftmaxGrad(float* dE_dy_l, float* y_l, float* dE_dx_l, int numCases, int numOut) {
    const int tx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified block indexing
    const int ty = blockIdx.y * blockDim.y + threadIdx.y; // Simplified block indexing
    const int tidx = ty * numCases + tx;

    if (ty < numOut && tx < numCases) {
        float v = 0.0f;
        float y_cache = y_l[tidx]; // Cache y_l for repeated access
        for (int j = 0; j < numOut; j++) {
            float dE_dy_cache = dE_dy_l[j * numCases + tx]; // Cache dE_dy_l for repeated access
            float y_comparison = (j == ty) ? 1.0f : y_l[j * numCases + tx]; // Avoid branching with ternary
            v += dE_dy_cache * (y_comparison - y_l[j * numCases + tx]);
        }
        dE_dx_l[tidx] = v * y_cache; // Use cached value
    }
}