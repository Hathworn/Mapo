#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate the global index for the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread index doesn't exceed array size
    if (i < blockDim.x * gridDim.x) {
        c[i] = a[i] + b[i];
    }
}