#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void knapsackKernel(int *profits, int *weights, int *input_f, int *output_f, int capacity, int c_min, int k) {

    int c = blockIdx.x * blockDim.x + threadIdx.x; // Generalized for different block sizes
    
    // Exit if not within valid capacity range
    if (c < c_min || c > capacity) return;

    int weight = weights[k - 1];
    int profit = profits[k - 1];
    
    if (c >= weight && input_f[c] < input_f[c - weight] + profit) {
        output_f[c] = input_f[c - weight] + profit; // Update only if the new profit is higher
    } else {
        output_f[c] = input_f[c]; // Retain the current profit otherwise
    }
}