#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void knapsackKernel(int *profits, int *weights, int *input_f, int *output_f, int capacity, int c_min, int k) {
    int c = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim for flexibility
    if (c < c_min || c > capacity) return;

    // Use shared memory for caching frequently accessed data
    __shared__ int sharedInput[512];
    sharedInput[threadIdx.x] = input_f[c];

    __syncthreads(); // Ensure shared memory is populated

    int weight = weights[k - 1];
    int profit = profits[k - 1];
    if (c >= weight) { // Ensure valid access to input_f array
        int candidate = input_f[c - weight] + profit;
        if (sharedInput[threadIdx.x] < candidate) {
            output_f[c] = candidate;
        } else {
            output_f[c] = sharedInput[threadIdx.x];
        }
    } else {
        output_f[c] = sharedInput[threadIdx.x]; // Default to current value if c < weight
    }
}