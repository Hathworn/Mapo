#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diffuseProject_k(float2 *vx, float2 *vy, int dx, int dy, float dt, float visc, int lb) {
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;

    if (gtidx < dx) {
        for (int p = 0; p < lb; p++) {
            int fi = gtidy + p;
            if (fi < dy) {
                int fj = fi * dx + gtidx;
                float2 xterm = vx[fj];
                float2 yterm = vy[fj];

                int iix = gtidx;
                int iiy = (fi > dy / 2) ? (fi - dy) : fi;

                float kk = (float)(iix * iix + iiy * iiy);  // k^2
                float diff = 1.f / (1.f + visc * dt * kk);
                xterm.x *= diff; xterm.y *= diff;
                yterm.x *= diff; yterm.y *= diff;

                // Velocity projection
                if (kk > 0.f) {
                    float rkk = 1.f / kk;
                    float rkp = (iix * xterm.x + iiy * yterm.x);
                    float ikp = (iix * xterm.y + iiy * yterm.y);
                    // Subtract projection component
                    xterm.x -= rkk * rkp * iix;
                    xterm.y -= rkk * ikp * iix;
                    yterm.x -= rkk * rkp * iiy;
                    yterm.y -= rkk * ikp * iiy;
                }

                vx[fj] = xterm;
                vy[fj] = yterm;
            }
        }
    }
}