#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void advectParticles_OGL(float2 *part, float2 *v, int dx, int dy, float dt, int lb, size_t pitch) {
    
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;
    
    // Use shared memory for particles
    extern __shared__ float2 shared_v[];

    // Pre-calculate metrics for memory access
    int threadsPerGroup = blockDim.x;
    
    // Inside domain check
    if (gtidx < dx) {
        for (p = 0; p < lb; p++) {
            int fi = gtidy + p;
            // Inside domain in Y check
            if (fi < dy) {
                int fj = fi * dx + gtidx;
                float2 pterm = part[fj];

                // Access velocity field
                int xvi = ((int)(pterm.x * dx)) % dx;
                int yvi = ((int)(pterm.y * dy)) % dy;
                shared_v[threadIdx.x] = *((float2*)((char*)v + yvi * pitch) + xvi);

                // Synchronize threads within this block to ensure all have loaded velocities
                __syncthreads();

                float2 vterm = shared_v[threadIdx.x];

                // Integrate particle position
                pterm.x += dt * vterm.x;
                pterm.x = fmodf(pterm.x + 1.0f, 1.0f);
                pterm.y += dt * vterm.y;
                pterm.y = fmodf(pterm.y + 1.0f, 1.0f);

                // Write back updated particle position
                part[fj] = pterm;
                
                // Synchronize to ensure all updates are complete before next loop iteration
                __syncthreads();
            }
        }
    }
}