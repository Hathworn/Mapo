#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateVelocity_k(float2 *v, float *vx, float *vy, int dx, int pdx, int dy, int lb, size_t pitch) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidyBase = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    float scale = 1.f / (dx * dy);

    // Only proceed if thread is within domain bounds in X
    if (gtidx < dx) {
        for (int p = 0; p < lb; p++) {
            int gtidy = gtidyBase + p;

            // Only proceed if thread is within domain bounds in Y
            if (gtidy < dy) {
                int fjr = gtidy * pdx + gtidx;
                
                // Load velocity components and scale
                float2 nvterm;
                nvterm.x = vx[fjr] * scale;
                nvterm.y = vy[fjr] * scale;

                // Update velocity vector
                float2 *fj = (float2*)((char*)v + gtidy * pitch) + gtidx;
                *fj = nvterm;
            }
        }
    }
}