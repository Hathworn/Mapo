#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    int local_factor = *factor; // Cache factor in a register
    int local_value = g_data[idx]; // Load value to a register for reuse

    for (int i = 0; i < num_iterations; i++)
    {
        local_value += local_factor; // Use register values in the loop
    }

    g_data[idx] = local_value; // Write the result back to global memory once
}