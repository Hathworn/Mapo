#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square(int *array, int arrayCount)
{
    extern __shared__ int dynamicSmem[];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use shared memory for frequently accessed data
    if (idx < arrayCount) {
        dynamicSmem[threadIdx.x] = array[idx];
        __syncthreads();
        
        dynamicSmem[threadIdx.x] *= dynamicSmem[threadIdx.x];
        
        __syncthreads();
        array[idx] = dynamicSmem[threadIdx.x];
    }
}