#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAdd(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Use shared memory for better memory access patterns in coalesced reads
    extern __shared__ float sdata[];

    if (i < N) {
        // Load A and B elements into shared memory
        sdata[i] = A[i];
        __syncthreads();

        // Perform the addition
        sdata[i] += B[i];
        __syncthreads();

        // Store the result back to global memory
        C[i] = sdata[i];
    }
}