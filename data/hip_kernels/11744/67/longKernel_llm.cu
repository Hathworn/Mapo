#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void longKernel(float *data, int N, float value) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if thread index is within bounds
    if (idx < N) {
        data[idx] += value;
    }
}