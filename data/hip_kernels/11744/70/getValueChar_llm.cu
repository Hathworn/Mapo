#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void getValueChar(char *outdata, char *indata) {
    // Use thread index to allow parallel processing
    int idx = threadIdx.x;
    outdata[idx] = indata[idx] + 3;
}