#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getValue(float *outdata, float *indata) {
    // Use thread index to allow for parallel execution
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < 1) {
        outdata[idx] = indata[idx] + 3.0f;
    }
}