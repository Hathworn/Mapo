#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void floattoint(int *out, float *in) {
    // Use shared memory for potential performance improvement
    __shared__ int shared_out[1];
    __shared__ float shared_in[1];

    // Load data into shared memory
    shared_in[0] = in[0];
    __syncthreads();

    // Efficiently convert float to int
    shared_out[0] = *(int *)&shared_in[0];
    __syncthreads();

    // Store result from shared memory to global memory
    out[0] = shared_out[0];
}