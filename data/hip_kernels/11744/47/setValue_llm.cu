#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setValue(float *data, int idx, float value) {
    // Use blockIdx.x to set value, avoiding warp divergence
    if(threadIdx.x + blockIdx.x * blockDim.x == idx) {
        data[idx] = value;
    }
}