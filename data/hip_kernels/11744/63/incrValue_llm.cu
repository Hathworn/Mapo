#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void incrValue(float *data, int idx, float value) {
    // Use atomicAdd to handle potential race conditions
    atomicAdd(&data[idx], value);
}