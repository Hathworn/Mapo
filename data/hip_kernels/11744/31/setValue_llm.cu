#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setValue(float *data, int idx, float value) {
    // Only one thread in the block is needed, we should ensure it's executed once per block
    if(blockIdx.x * blockDim.x + threadIdx.x == 0) {
        data[idx] = value;
    }
}