#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use CUDA parallelization to optimize the loop inside the kernel
__global__ void longKernel(float *data, int N, float value) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N) {
        data[i] += value;  // Perform the addition in parallel
    }
}