#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inttofloat(float *out, int *in) {
    int idx = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x; // Calculate global thread index
    out[idx] = __int_as_float(in[idx]); // Efficient int-to-float conversion
}