#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getValue(float4 *outdata, float *indata) {
    // Use thread indexing for parallel processing
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx == 0) {
        float4 my4 = make_float4(indata[0], indata[3], indata[1], indata[2]);
        outdata[0] = my4;
    }
}