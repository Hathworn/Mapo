#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getValue(float *outdata, float *indata) {
    // Utilize block and thread indices for parallel processing
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // Guard against excess threads
    if (idx == 0) {
        outdata[idx] = indata[idx] == 0 ? 3.0f : 2.0f;
    }
}