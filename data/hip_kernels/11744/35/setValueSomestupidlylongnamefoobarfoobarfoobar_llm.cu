#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void setValue(float *data, int idx, float value) {
    data[idx] = value;
}

__device__ float bar(float a, float b) {
    return a + b;
}

__global__ void setValueOptimized(float *data, int idx, float value) {
    // Eliminate unnecessary thread check by using a single thread block
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        setValue(data, idx, value);
    }
}