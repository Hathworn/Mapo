#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void longKernel(float *data, int N, float value) {
    // Calculate unique thread index in global memory
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we do not go out of bounds
    if (idx < N) {
        data[idx] += value;
    }
}