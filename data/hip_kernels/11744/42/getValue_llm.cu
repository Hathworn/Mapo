#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getValue(float *outdata, const float *indata) {
    // Use blockIdx.x and threadIdx.x to allow parallel processing
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure that outdata[0] is updated using the first thread
    if (idx == 0) {
        outdata[0] = (*indata == 0) ? 3.0f : 2.0f;
    }
}