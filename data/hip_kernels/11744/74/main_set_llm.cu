#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void main_set(float *data, float val) {
    // Calculate global index to handle larger arrays
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Assign value if within bounds (assuming proper data length allocation)
    data[idx] = val;
}