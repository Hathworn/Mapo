#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void wipe(int *buffer, int length) {
    // Calculate global thread index
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Use stride loop for parallel execution
    for (int i = globalIdx; i < length; i += stride) {
        buffer[i] = -1;
    }
}