#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function by removing condition and using block-level synchronization
__global__ void setValue(float *data, int idx, float value) {
    if(blockIdx.x * blockDim.x + threadIdx.x == idx) {
        data[idx] = value;
    }
}