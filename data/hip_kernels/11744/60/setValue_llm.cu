#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setValue(float *data, int idx, float value) {
    int globalIdx = blockDim.x * blockIdx.x + threadIdx.x; // Calculate global index
    if (globalIdx == idx) { // Check if current thread corresponds to the target index
        data[globalIdx] = value; // Set value
    }
}