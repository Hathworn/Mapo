#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

__global__ void setValue(float *data, int idx, float value) {
    // Optimize by using atomic operation to avoid potential race conditions
    if(threadIdx.x == 0 && idx < blockDim.x * gridDim.x) {
        atomicExch(&data[idx], value);
    }
}