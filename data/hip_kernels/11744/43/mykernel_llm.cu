#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mykernel(float *d1, float *d2, float *d3, float *d4, float *d5) {
    // Use blockDim to ensure only one thread executes the conditional
    if(blockIdx.x == 0 && threadIdx.x == 0) {
        // Store constant value in shared variable to reduce redundancy
        const float value = 123.0f;
        d1[0] = value;
        d2[0] = value;
        d3[0] = value;
        d4[0] = value;
        d5[0] = value;
    }
}