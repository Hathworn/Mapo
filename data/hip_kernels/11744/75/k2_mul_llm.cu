#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void k2_mul(float *data, float val) {
// Use blockDim.x for better occupancy and handling larger data sizes
int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < N) { // Ensure thread doesn't access out-of-bounds
    data[idx] *= val;
}
}