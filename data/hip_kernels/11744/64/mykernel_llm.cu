#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mykernel(int *int1, float *f1, int *int2) {
    // Load int value and store it in float array using reinterpret_cast for better readability
    f1[0] = __int_as_float(int1[0]);

    // Load float value and store it in int array using reinterpret_cast for better readability
    int2[0] = __float_as_int(f1[0]);
}