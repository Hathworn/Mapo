#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void longKernel(float *data, int N, float value) {
    // Calculate the global index for each thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use the global index to directly access and modify data
    if (idx < N) {
        data[idx] += value;
    }
}