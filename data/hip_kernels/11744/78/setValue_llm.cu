#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setValue(float *data, int idx, float value) {
    // Use blockIdx.x to allow setting multiple values in parallel
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x; // Compute global index
    if(globalIdx == idx) {
        data[idx] = value;
    }
}