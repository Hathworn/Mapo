#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getValue(float *outdata, float *indata) {
    // Using blockIdx.x and threadIdx.x for indexing allows this kernel to handle larger data arrays.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) { // Limited work to only one thread as original logic implies operation on indata[0].
        outdata[0] = indata[0] + 3.0f;
    }
}