#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getValue(float *outdata, float *indata) {
    // Use ternary operator outside of array access for better readability
    outdata[0] = (indata[0] == 0.0f) ? 3.0f : 2.0f; // Correct the indata check
}