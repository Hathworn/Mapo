#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rmspropKernel(int numberIterations, int* parameterIndices, int* counts, int dimension, float* parameters, float* gradient, float learningRate, float decay, float oneMinusDecay, float epsilon, float* accumulation) {

    int updateIndex = blockIdx.x;
    int parameterIndex = parameterIndices[updateIndex];
    int count = counts[updateIndex];

    // Check if update is necessary
    if (parameterIndex != -1 && count > 0) {

        // Use shared memory for faster access
        extern __shared__ float sharedGrad[];

        float scalingFactor = 1.0f / (float)count;
        int firstParameterEntryIndex = parameterIndex * dimension;
        int startEntryIndex = (blockIdx.y * blockDim.x + threadIdx.x) * numberIterations;
        int startParameterEntryIndex = firstParameterEntryIndex + startEntryIndex;
        int startGradientEntryIndex = updateIndex * dimension + startEntryIndex;
        int exclusiveEndParameterEntryIndex = min(startParameterEntryIndex + numberIterations, firstParameterEntryIndex + dimension);

        // Load gradient to shared memory; assuming numberIterations <= blockDim.x for simplicity
        if (threadIdx.x < numberIterations) {
            sharedGrad[threadIdx.x] = gradient[startGradientEntryIndex + threadIdx.x];
        }
        __syncthreads(); // Ensure all threads have loaded the gradients

        for (int offset = 0; offset < numberIterations; offset++) {
            int parameterEntryIndex = startParameterEntryIndex + offset;
            if (parameterEntryIndex < exclusiveEndParameterEntryIndex) {
                float scaledDerivative = scalingFactor * sharedGrad[offset]; // Use shared memory
                float updatedAccumulation = decay * accumulation[parameterEntryIndex] + oneMinusDecay * (scaledDerivative * scaledDerivative);
                accumulation[parameterEntryIndex] = updatedAccumulation;

                float adaptiveLearningRate = learningRate / sqrtf(updatedAccumulation + epsilon);
                float update = -adaptiveLearningRate * scaledDerivative;
                parameters[parameterEntryIndex] += update;
            }
        }
    }
}