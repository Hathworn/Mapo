#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backwardMaxPoolingKernel(int batchSize, int* lengths, int numberEntriesPerInstance, int numberRows, int* maxIndices, float* chain, float* result) {
    int indexInstance = blockIdx.x;
    int indexRow = blockIdx.y;
    int indexColumn = threadIdx.x;

    // Pre-compute reused values outside the conditional
    int startInstanceWithinBatch = indexInstance * numberEntriesPerInstance;
    int startColumnWithinInstance = indexColumn * numberRows;
    int indexEntryWithinBatch = startInstanceWithinBatch + startColumnWithinInstance + indexRow;

    if (indexInstance < batchSize) {
        int length = lengths[indexInstance];

        // Simplify conditional to check bounds first
        if (indexColumn < length) {
            int maxIndex = maxIndices[indexInstance * numberRows + indexRow];

            // Assign result based on whether it's a max index
            result[indexEntryWithinBatch] = (indexEntryWithinBatch == maxIndex) 
                                            ? chain[indexInstance * numberRows + indexRow]
                                            : 0.0f;
        } else {
            result[indexEntryWithinBatch] = nanf("NaN");
        }
    } else {
        result[indexEntryWithinBatch] = nanf("NaN");
    }
}