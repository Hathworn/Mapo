#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(float* A, float* B, float* C, int numberIterations, int size) {

    // Calculate global thread index
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Compute starting index for this thread
    int start = globalIndex * numberIterations;
    
    // Use shared memory for a section of A and B
    extern __shared__ float sharedMem[];
    float* sharedA = sharedMem;
    float* sharedB = sharedMem + blockDim.x * numberIterations;

    // Load data into shared memory
    for (int i = 0; i < numberIterations && (start + i) < size; ++i) {
        int idx = start + i;
        sharedA[threadIdx.x * numberIterations + i] = A[idx];
        sharedB[threadIdx.x * numberIterations + i] = B[idx];
    }

    // Synchronize to make sure data is loaded before using it
    __syncthreads();
    
    // Perform addition
    for (int i = 0; i < numberIterations && (start + i) < size; ++i) {
        int idx = start + i;
        C[idx] = sharedA[threadIdx.x * numberIterations + i] + sharedB[threadIdx.x * numberIterations + i];
    }
}