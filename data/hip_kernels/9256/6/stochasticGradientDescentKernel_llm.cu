#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stochasticGradientDescentKernel (int numberIterations, int* parameterIndices, int* counts, int dimension, float* parameters, float* gradient, float learningRate) {

    int updateIndex = blockIdx.x;
    int parameterIndex = parameterIndices[updateIndex];
    int count = counts[updateIndex];

    if(parameterIndex != -1 && count > 0) {

        float scalingFactor = 1.0f / (float)count;

        int startEntryIndex = (blockIdx.y * blockDim.x + threadIdx.x) * numberIterations;

        int firstParameterEntryIndex = parameterIndex * dimension;
        int startParameterEntryIndex = firstParameterEntryIndex + startEntryIndex;
        int startGradientEntryIndex = updateIndex * dimension + startEntryIndex;

        int exclusiveEndParameterEntryIndex = min(startParameterEntryIndex + numberIterations, firstParameterEntryIndex + dimension);

        // Unroll the loop for better warp utilization
        for(int parameterEntryIndex = startParameterEntryIndex, gradientEntryIndex = startGradientEntryIndex;
            parameterEntryIndex < exclusiveEndParameterEntryIndex; parameterEntryIndex++, gradientEntryIndex++) {
            
            float scaledDerivative = scalingFactor * gradient[gradientEntryIndex];
            parameters[parameterEntryIndex] -= learningRate * scaledDerivative;
        }
    }
}