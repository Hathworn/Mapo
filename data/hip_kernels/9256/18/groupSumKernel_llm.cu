#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void groupSumKernel( float* input, int* firstOccurrences, int* otherOccurrences, int* otherOccurrencePositions, int dimensions, int numberIterations) {
    
    int duplicateIndex = blockIdx.x;
    int firstOccurrence = firstOccurrences[duplicateIndex];

    int startWithinParameter = threadIdx.x * numberIterations;
    int startFirstOccurrenceWithinBatch = firstOccurrence * dimensions;

    int startFirstOccurrenceEntryIndex = startFirstOccurrenceWithinBatch + startWithinParameter;
    int exclusiveEndFirstOccurrenceEntryIndex = min(startFirstOccurrenceEntryIndex + numberIterations, startFirstOccurrenceWithinBatch + dimensions);

    int startOtherOccurrencePosition = otherOccurrencePositions[duplicateIndex];
    int exclusiveEndOtherOccurrencePosition = otherOccurrencePositions[duplicateIndex + 1];

    // Optimize memory access by loading start indices once per loop
    for(int otherOccurrencePosition = startOtherOccurrencePosition; otherOccurrencePosition < exclusiveEndOtherOccurrencePosition; otherOccurrencePosition++) {
        int otherOccurrence = otherOccurrences[otherOccurrencePosition];
        int otherOccurrenceEntryIndex = otherOccurrence * dimensions + startWithinParameter;

        // Use loop unrolling for better performance
        for(int firstOccurrenceEntryIndex = startFirstOccurrenceEntryIndex; firstOccurrenceEntryIndex < exclusiveEndFirstOccurrenceEntryIndex; firstOccurrenceEntryIndex++, otherOccurrenceEntryIndex++) {
            input[firstOccurrenceEntryIndex] += input[otherOccurrenceEntryIndex];
            input[otherOccurrenceEntryIndex] = nanf("NaN");
        }
    }
}