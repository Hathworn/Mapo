#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumKernel(float* input, float* result, int batchSize, int numberEntries, int numberIterations) {

    int startEntry = (blockIdx.x * blockDim.x + threadIdx.x) * numberIterations;
    int exclusiveEndEntry = min(startEntry + numberIterations, numberEntries);

    // Use registers for the entry variable to reduce memory access
    for (int entryIndex = startEntry; entryIndex < exclusiveEndEntry; entryIndex++) {
        
        float entry = 0.0f;

        // Use shared memory to cache input data within a block for faster access
        __shared__ float sharedInput[1024]; // Assumes batchSize * numberEntries <= 1024
        for (int instanceIndex = threadIdx.x; instanceIndex < batchSize; instanceIndex += blockDim.x) {
            sharedInput[instanceIndex * numberEntries + entryIndex] = input[instanceIndex * numberEntries + entryIndex];
        }
        __syncthreads();

        for (int instanceIndex = 0; instanceIndex < batchSize; instanceIndex++) {
            entry += sharedInput[instanceIndex * numberEntries + entryIndex];
        }

        result[entryIndex] = entry;
    }

}