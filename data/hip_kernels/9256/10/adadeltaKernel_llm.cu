#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adadeltaKernel (int numberIterations, int* parameterIndices, int* counts, int dimension, float* parameters, float* gradient, float decay, float oneMinusDecay, float epsilon, float* gradientAccumulation, float* updateAccumulation) {

    int updateIndex = blockIdx.x;
    int parameterIndex = parameterIndices[updateIndex];
    int count = counts[updateIndex];

    if (parameterIndex != -1 && count > 0) {

        // Precompute scaling factor outside the loop
        float scalingFactor = 1.0f / (float)count;

        int startEntryIndex = (blockIdx.y * blockDim.x + threadIdx.x) * numberIterations;

        int firstParameterEntryIndex = parameterIndex * dimension;
        int startParameterEntryIndex = firstParameterEntryIndex + startEntryIndex;
        int startGradientEntryIndex = updateIndex * dimension + startEntryIndex;

        int exclusiveEndParameterEntryIndex = min(startParameterEntryIndex + numberIterations, firstParameterEntryIndex + dimension);

        for (int parameterEntryIndex = startParameterEntryIndex, gradientEntryIndex = startGradientEntryIndex; 
             parameterEntryIndex < exclusiveEndParameterEntryIndex; 
             ++parameterEntryIndex, ++gradientEntryIndex) {
        
            // Load gradient once to reduce redundant memory accesses
            float scaledDerivative = scalingFactor * gradient[gradientEntryIndex];

            float newGradientAccumulation = decay * gradientAccumulation[parameterEntryIndex] + oneMinusDecay * (scaledDerivative * scaledDerivative);
            gradientAccumulation[parameterEntryIndex] = newGradientAccumulation;

            float rootMeanSquaredOfDerivatives = rsqrtf(newGradientAccumulation + epsilon); // Use rsqrt for better performance

            float pastUpdateAccumulation = updateAccumulation[parameterEntryIndex];
            float rootMeanSquaredOfPastUpdates = sqrtf(pastUpdateAccumulation + epsilon);

            float learningRate = rootMeanSquaredOfPastUpdates * rootMeanSquaredOfDerivatives; // Avoid division for performance

            float update = -learningRate * scaledDerivative;

            updateAccumulation[parameterEntryIndex] = decay * pastUpdateAccumulation + oneMinusDecay * (update * update);

            parameters[parameterEntryIndex] += update;
        }
    }
}