#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiClassTestingKernel(int batchStart, int numberRows, int numberColumns, int numberEntriesPerInstance, float* predictions, float* targets, int* result) {

    int indexInstance = blockIdx.x;
    int instanceStart = indexInstance * numberEntriesPerInstance;

    // Cache batchStart + indexInstance to avoid recalculating
    int resultIndex = batchStart + indexInstance;

    bool allMatched = true; // Flag to determine if all columns matched

    for (int indexColumn = 0; indexColumn < numberColumns; indexColumn++) {
        int instanceColumnStart = instanceStart + indexColumn * numberRows;
        float maximumPrediction = 0;
        int maximumPredictionIndex = -1;

        float maximumTarget = 0;
        int maximumTargetIndex = -1;

        // Unroll loop with pragma for potential performance gains
        #pragma unroll
        for (int indexEntryOffset = 0; indexEntryOffset < numberRows; indexEntryOffset++) {
            int indexEntry = instanceColumnStart + indexEntryOffset;
            float prediction = predictions[indexEntry];
            float target = targets[indexEntry];

            if (prediction > maximumPrediction) {
                maximumPrediction = prediction;
                maximumPredictionIndex = indexEntryOffset; // Use offset index for comparison
            }

            if (target > maximumTarget) {
                maximumTarget = target;
                maximumTargetIndex = indexEntryOffset;
            }
        }

        // Check if predictions and targets match, break early if not matched
        if (maximumPredictionIndex != maximumTargetIndex) {
            allMatched = false;
            break;
        }
    }

    result[resultIndex] = allMatched ? 1 : 0; // Set result based on match flag
}