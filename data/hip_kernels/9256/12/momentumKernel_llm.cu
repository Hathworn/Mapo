#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void momentumKernel(int numberIterations, int* parameterIndices, int* counts, int dimension, float* parameters, float* gradient, float learningRate, float momentum, float* history) {

    int updateIndex = blockIdx.x;
    int parameterIndex = parameterIndices[updateIndex];
    int count = counts[updateIndex];

    if (parameterIndex != -1 && count > 0) {
        float scalingFactor = 1.0f / count;

        int startEntryIndex = (blockIdx.y * blockDim.x + threadIdx.x) * numberIterations;

        int firstParameterEntryIndex = parameterIndex * dimension;
        int startParameterEntryIndex = firstParameterEntryIndex + startEntryIndex;
        int startGradientEntryIndex = updateIndex * dimension + startEntryIndex;

        int exclusiveEndParameterEntryIndex = min(startParameterEntryIndex + numberIterations, firstParameterEntryIndex + dimension);

        // Use stride to handle parameter and gradient updates for multiple steps in one iteration 
        for (int i = startEntryIndex; i < exclusiveEndParameterEntryIndex; i += numberIterations) {
            float derivative = gradient[startGradientEntryIndex + i - startEntryIndex]; // Offset index calculation
            float scaledDerivative = scalingFactor * derivative;

            float update = momentum * history[startParameterEntryIndex + i - startEntryIndex] - learningRate * scaledDerivative;

            history[startParameterEntryIndex + i - startEntryIndex] = update;
            parameters[startParameterEntryIndex + i - startEntryIndex] += update;
        }
    }
}