#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binaryTestingKernel ( int batchStart, int length, float* predictions, float* targets, int* result) {
    int instanceStart = batchStart + blockIdx.x * length;
    int indexEntry = instanceStart + threadIdx.x;
    
    // Use thread-level parallelism and guard for out-of-bounds
    if (indexEntry < instanceStart + length) {
        float prediction = predictions[indexEntry];
        float target = targets[indexEntry];
        result[indexEntry] = (prediction < 0.5 && target == 0.0) || (prediction >= 0.5 && target == 1.0);
    }
}