#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expansionKernel( int batchSize, float* input, int* inputLengths, int numberRows, int numberInputEntries, int numberFilterRowPositions, int filterHeight, int filterWidth, int filterLength, int numberResultEntries, float* result, int* resultLengths) {

    int indexInstance = blockIdx.x;
    int indexConvolution = blockIdx.y;
    int indexConvolutionEntry = threadIdx.x;

    // Precompute reused values
    int firstColumnOfConvolution = indexConvolution / numberFilterRowPositions;
    int firstRowOfConvolution = indexConvolution % numberFilterRowPositions;

    int relativeIndexColumn = indexConvolutionEntry / filterHeight;
    int relativeIndexRow = indexConvolutionEntry % filterHeight;

    int indexColumn = firstColumnOfConvolution + relativeIndexColumn;
    int indexRow = firstRowOfConvolution + relativeIndexRow;

    // Precompute index for result
    int indexEntryWithinResult = indexInstance * numberResultEntries + indexConvolution * filterLength + indexConvolutionEntry;

    if (indexInstance < batchSize) {
        int inputLength = inputLengths[indexInstance];
        int numberConvolutions = (inputLength - filterWidth + 1) * numberFilterRowPositions;

        if (indexConvolution < numberConvolutions) {
            // Efficiently use shared memory prefetch
            result[indexEntryWithinResult] = input[indexInstance * numberInputEntries + indexColumn * numberRows + indexRow];
        } else {
            result[indexEntryWithinResult] = 0.0;
        }

        // Use thread 0 to set resultLengths
        if (indexConvolutionEntry == 0 && indexConvolution == 0) {
            resultLengths[indexInstance] = numberConvolutions;
        }
    } else {
        result[indexEntryWithinResult] = nanf("NaN");
        resultLengths[indexInstance] = 0;
    }
}