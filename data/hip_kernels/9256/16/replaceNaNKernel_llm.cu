#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void replaceNaNKernel(int numberEntriesPerInstance, int numberIterations, float* source, float* destination) {

    int instanceStart = blockIdx.x * numberEntriesPerInstance;
    int baseIndex = instanceStart + (blockIdx.y * blockDim.x + threadIdx.x) * numberIterations;

    // Loop unrolling for better performance
    #pragma unroll
    for (int i = 0; i < numberIterations; i++) {
        int index = baseIndex + i;
        if (index < instanceStart + numberEntriesPerInstance) {
            float currentValue = source[index];
            destination[index] = __builtin_isnan(currentValue) ? 0.0f : currentValue;
        }
    }
}