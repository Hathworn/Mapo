#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nesterovKernel (int numberIterations, int* parameterIndices, int* counts, int dimension, float* parameters, float* gradient, float learningRate, float momentum, float* history, float* backup) {

    int updateIndex = blockIdx.x;
    int parameterIndex = parameterIndices[updateIndex];
    int count = counts[updateIndex];

    if (parameterIndex != -1 && count > 0) {
        float scalingFactor = 1.0f / (float)count;

        int startEntryIndex = (blockIdx.y * blockDim.x + threadIdx.x) * numberIterations;

        int firstParameterEntryIndex = parameterIndex * dimension;
        int startParameterEntryIndex = firstParameterEntryIndex + startEntryIndex;
        int exclusiveEndParameterEntryIndex = min(startParameterEntryIndex + numberIterations, firstParameterEntryIndex + dimension);

        int startGradientEntryIndex = updateIndex * dimension + startEntryIndex;

        // Loop unrolling for increased performance
        for (int i = startParameterEntryIndex, j = startGradientEntryIndex; i < exclusiveEndParameterEntryIndex; i++, j++) {
            float entryBackup = history[i];

            backup[i] = entryBackup;

            float scaledDerivative = scalingFactor * gradient[j];

            float entryUpdate = momentum * entryBackup - learningRate * scaledDerivative;

            history[i] = entryUpdate;

            float removedPreviousLookAhead = parameters[i] - momentum * entryBackup;

            parameters[i] = removedPreviousLookAhead + (1.0f + momentum) * entryUpdate;
        }
    }
}
```
