#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adagradKernel (int numberIterations, int* parameterIndices, int* counts, int dimension, float* parameters, float* gradient, float learningRate, float* history, float epsilon) {

    int updateIndex = blockIdx.x;
    int parameterIndex = parameterIndices[updateIndex];
    int count = counts[updateIndex];

    if (parameterIndex != -1 && count > 0) {

        float scalingFactor = 1.0f / (float)count;

        int startEntryIndex = (blockIdx.y * blockDim.x + threadIdx.x) * numberIterations;

        int firstParameterEntryIndex = parameterIndex * dimension;
        int startParameterEntryIndex = firstParameterEntryIndex + startEntryIndex;
        int startGradientEntryIndex = updateIndex * dimension + startEntryIndex;

        int exclusiveEndParameterEntryIndex = min(startParameterEntryIndex + numberIterations, firstParameterEntryIndex + dimension);

        // Unroll loop for performance
        for (int i = startParameterEntryIndex, j = startGradientEntryIndex; i < exclusiveEndParameterEntryIndex; ++i, ++j) {
            float scaledDerivative = scalingFactor * gradient[j];
            float updatedHistory = history[i] + scaledDerivative * scaledDerivative;
            history[i] = updatedHistory;
            float adaptedLearningRate = learningRate / (sqrtf(updatedHistory) + epsilon);
            float update = adaptedLearningRate * scaledDerivative;
            parameters[i] -= update;
        }
    }
}