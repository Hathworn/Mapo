#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize by using a single thread to perform division
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}