#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
    // Optimize by reducing global memory accesses and increasing parallelism using block and grid dimensions.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N * gridDim.x) {
        int row = idx / N;
        int col = idx % N;
        output[idx] = K_A_B - row_sum[row] - col_sum[col];
    }
}