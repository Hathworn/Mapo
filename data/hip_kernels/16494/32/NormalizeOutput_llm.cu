#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Optimize using shared memory for fewer global memory accesses
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    extern __shared__ int temp[];
    
    if (idx < num_elements) {
        int local_idx = threadIdx.x;
        temp[local_idx] = original[idx];
        __syncthreads(); // Ensure all threads have loaded the data

        to_normalize[idx * 3] = batch_index;
        to_normalize[idx * 3 + 1] = class_index;
        to_normalize[idx * 3 + 2] = static_cast<int64_t>(temp[local_idx]);
    }
}