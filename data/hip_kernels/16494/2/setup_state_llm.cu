#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Optimize to initialize hiprand state in parallel using threadIdx
    int idx = hipThreadIdx_x;
    hiprand_init(seed, idx, 0, &state[idx]);
}