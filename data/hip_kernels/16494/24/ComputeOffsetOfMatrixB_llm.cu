#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Calculate unique thread index in grid
    int32_t global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within bounds
    if (global_idx < N) {
        output[global_idx] = -row_sum[blockIdx.x];
    }
}