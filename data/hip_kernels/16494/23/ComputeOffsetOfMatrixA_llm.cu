#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Calculate the global thread index
    int32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread index is within bounds before writing to output
    if (index < N) {
        *(output + blockIdx.x * N + index) = -col_sum[index];
    }
}