#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PadMatrixInLeadingDimensionKernel(const int8_t* src, int8_t* dst, int col_src, int col_dst) {
    // Calculate the starting index for the thread in the matrix
    int32_t startIdx = threadIdx.x + blockIdx.x * blockDim.x;
    int32_t stride = blockDim.x * gridDim.x; // Total threads in the grid
    for (int32_t i = startIdx; i < col_src; i += stride) {
        *(dst + blockIdx.x * col_dst + i) = *(src + blockIdx.x * col_src + i);
    }
}
```
