#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ size_t gpu_fieldn_index(unsigned int x, unsigned int y, unsigned int d) {
    return (NX * (NY * (d - 1) + y) + x);
}

__device__ __forceinline__ size_t gpu_field0_index(unsigned int x, unsigned int y) {
    return NX * y + x;
}

__global__ void gpu_bc_charge(double *h0, double *h1, double *h2) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y;

    // Use shared memory to reduce repeated calculations
    if (y == 0) {
        // Pre-compute repeated multipliers
        double multi0c = 2.0 * charge0 * w0;
        double multisc = 2.0 * charge0 * ws;
        double multidc = 2.0 * charge0 * wd;

        // Cache repeated index calculations
        size_t base_index = gpu_field0_index(x, 0);
        size_t fieldn_index0 = gpu_fieldn_index(x, 0, 1);
        size_t fieldn_index1 = gpu_fieldn_index(x, 0, 2);
        size_t fieldn_index2 = gpu_fieldn_index(x, 0, 3);
        size_t fieldn_index3 = gpu_fieldn_index(x, 0, 4);
        size_t fieldn_index4 = gpu_fieldn_index(x, 0, 5);
        size_t fieldn_index5 = gpu_fieldn_index(x, 0, 6);
        size_t fieldn_index6 = gpu_fieldn_index(x, 0, 7);
        size_t fieldn_index7 = gpu_fieldn_index(x, 0, 8);

        // Read values once into local variables
        double ht1 = h2[fieldn_index0];
        double ht2 = h2[fieldn_index1];
        double ht3 = h2[fieldn_index2];
        double ht4 = h2[fieldn_index3];
        double ht5 = h2[fieldn_index4];
        double ht6 = h2[fieldn_index5];
        double ht7 = h2[fieldn_index6];
        double ht8 = h2[fieldn_index7];

        // Update h0/h1 arrays with the necessary calculations
        h0[base_index] = -h0[base_index] + multi0c;
        h1[fieldn_index2] = -ht1 + multisc;
        h1[fieldn_index3] = -ht2 + multisc;
        h1[fieldn_index0] = -ht3 + multisc;
        h1[fieldn_index1] = -ht4 + multisc;
        h1[fieldn_index6] = -ht5 + multidc;
        h1[fieldn_index7] = -ht6 + multidc;
        h1[fieldn_index4] = -ht7 + multidc;
        h1[fieldn_index5] = -ht8 + multidc;
    }
}