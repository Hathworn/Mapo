#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ size_t gpu_scalar_index(unsigned int x, unsigned int y)
{
    return NX * y + x;
}

__global__ void gpu_efield(double *fi, double *ex, double *ey)
{
    unsigned int y = blockIdx.y;
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit for threads outside the x boundary
    if (x >= NX) return;

    unsigned int xp1 = (x + 1) % NX;
    unsigned int yp1 = (y + 1) % NY;
    unsigned int xm1 = (NX + x - 1) % NX;
    unsigned int ym1 = (NY + y - 1) % NY;

    size_t idx = gpu_scalar_index(x, y);
    double phi  = fi[idx];
    double phiL = fi[gpu_scalar_index(xm1, y)];
    double phiR = fi[gpu_scalar_index(xp1, y)];
    double phiU = fi[gpu_scalar_index(x, yp1)];
    double phiD = fi[gpu_scalar_index(x, ym1)];

    // Compute electric field components
    ex[idx] = 0.5 * (phiL - phiR) / dx;
    ey[idx] = 0.5 * (phiD - phiU) / dy;
}