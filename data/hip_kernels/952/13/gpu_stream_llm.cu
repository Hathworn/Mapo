#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ size_t gpu_fieldn_index(unsigned int x, unsigned int y, unsigned int d)
{
    return (NX * (NY * (d - 1) + y) + x);
}

__global__ void gpu_stream(double *f0, double *f1, double *f2, double *h0, double *h1, double *h2)
{
    unsigned int y = blockIdx.y;
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Precompute indices to reduce redundant calculations
    unsigned int idx_xm1 = (NX + x - 1) % NX;
    unsigned int idx_xp1 = (x + 1) % NX;
    unsigned int idx_ym1 = (NY + y - 1) % NY;
    unsigned int idx_yp1 = (y + 1) % NY;

    size_t idx_base = gpu_fieldn_index(x, y, 0);

    // Use temporary variables to minimize repeated indexing and accesses
    double t_f1[8], t_h1[8], t_f2[8], t_h2[8];

    t_f2[0] = f2[idx_base + gpu_fieldn_index(idx_xm1, y, 1)];
    t_f2[1] = f2[idx_base + gpu_fieldn_index(x, idx_ym1, 2)];
    t_f2[2] = f2[idx_base + gpu_fieldn_index(idx_xp1, y, 3)];
    t_f2[3] = f2[idx_base + gpu_fieldn_index(x, idx_yp1, 4)];
    t_f2[4] = f2[idx_base + gpu_fieldn_index(idx_xm1, idx_ym1, 5)];
    t_f2[5] = f2[idx_base + gpu_fieldn_index(idx_xp1, idx_ym1, 6)];
    t_f2[6] = f2[idx_base + gpu_fieldn_index(idx_xp1, idx_yp1, 7)];
    t_f2[7] = f2[idx_base + gpu_fieldn_index(idx_xm1, idx_yp1, 8)];

    t_h2[0] = h2[idx_base + gpu_fieldn_index(idx_xm1, y, 1)];
    t_h2[1] = h2[idx_base + gpu_fieldn_index(x, idx_ym1, 2)];
    t_h2[2] = h2[idx_base + gpu_fieldn_index(idx_xp1, y, 3)];
    t_h2[3] = h2[idx_base + gpu_fieldn_index(x, idx_yp1, 4)];
    t_h2[4] = h2[idx_base + gpu_fieldn_index(idx_xm1, idx_ym1, 5)];
    t_h2[5] = h2[idx_base + gpu_fieldn_index(idx_xp1, idx_ym1, 6)];
    t_h2[6] = h2[idx_base + gpu_fieldn_index(idx_xp1, idx_yp1, 7)];
    t_h2[7] = h2[idx_base + gpu_fieldn_index(idx_xm1, idx_yp1, 8)];

    // Write from temporary variables to output arrays to improve locality
    for (int i = 0; i < 8; i++)
    {
        f1[idx_base + gpu_fieldn_index(x, y, i + 1)] = t_f2[i];
        h1[idx_base + gpu_fieldn_index(x, y, i + 1)] = t_h2[i];
    }
}