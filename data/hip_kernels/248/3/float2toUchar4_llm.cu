#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void float2toUchar4(float2 *inputImage, uchar4 *outputImage, int width, int height, int index)
{
    // Calculate global thread index more efficiently
    int offset = blockIdx.x * blockDim.x + threadIdx.x + 
                 (blockIdx.y * blockDim.y + threadIdx.y) * width;

    if (offset < width * height) // Ensure within bounds
    {
        float2 pixelf = inputImage[offset];
        float pixelfIndexed = (index == 0) ? pixelf.x : pixelf.y;

        unsigned char absPixel = (unsigned char)abs(pixelfIndexed);

        uchar4 pixel;
        // Set all channels using a single variable
        pixel.x = absPixel; 
        pixel.y = absPixel;
        pixel.z = absPixel; 
        pixel.w = absPixel;

        outputImage[offset] = pixel;
    }
}