#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kRgb2XYZ(uchar4* inputImg, float4* outputImg, int width, int height)
{
    // Optimize offset calculation using a single linear index
    int offset = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y 
                 + threadIdx.y * blockDim.x + threadIdx.x;

    // Ensure the thread is within image bounds
    if (offset >= width * height) return;

    uchar4 nPixel = inputImg[offset];

    float _b = (float)nPixel.x / 255.0f;
    float _g = (float)nPixel.y / 255.0f;
    float _r = (float)nPixel.z / 255.0f;

    // Utilize registers for computation
    float x = _r * 0.412453f + _g * 0.357580f + _b * 0.180423f;
    float y = _r * 0.212671f + _g * 0.715160f + _b * 0.072169f;
    float z = _r * 0.019334f + _g * 0.119193f + _b * 0.950227f;

    // Store results directly in float4
    outputImg[offset] = make_float4(x, y, z, 0.0f);
}