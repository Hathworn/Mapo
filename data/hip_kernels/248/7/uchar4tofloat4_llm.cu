#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uchar4tofloat4(uchar4 *inputImage, float4 *outputImage, int width, int height)
{
    // Calculate the unique thread index for the 2D grid
    int offsetX = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetY = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure within bounds
    if (offsetX < width && offsetY < height)
    {
        // Use a single step to calculate the offset
        int offset = offsetY * width + offsetX;

        // Get pixel
        uchar4 pixel = inputImage[offset];

        // Optimize float4 assignment
        outputImage[offset] = make_float4(pixel.x, pixel.y, pixel.z, pixel.w);
    }
}