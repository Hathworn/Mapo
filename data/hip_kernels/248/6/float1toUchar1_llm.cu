#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void float1toUchar1(float1 *inputImage, uchar1 *outputImage, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x; // Calculate x-coordinate uniquely for each thread
    int y = blockIdx.y * blockDim.y + threadIdx.y; // Calculate y-coordinate uniquely for each thread
    
    if(x < width && y < height) // Boundary check for image dimensions
    {
        int offset = y * width + x; // Calculate 1D offset from 2D coordinates

        float1 pixelf = inputImage[offset];
        uchar1 pixel;
        pixel.x = (unsigned char)pixelf.x;

        outputImage[offset] = pixel;
    }
}