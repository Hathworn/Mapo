#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kRgb2CIELab(uchar4* inputImg, float4* outputImg, int width, int height)
{
    // Calculate the offset using optimized block and thread indexing
    int offset = (blockIdx.y * blockDim.y + threadIdx.y) * width + blockIdx.x * blockDim.x + threadIdx.x;

    uchar4 nPixel = inputImg[offset];

    // Normalize the RGB values
    float _b = static_cast<float>(nPixel.x) / 255.0f;
    float _g = static_cast<float>(nPixel.y) / 255.0f;
    float _r = static_cast<float>(nPixel.z) / 255.0f;

    // Convert RGB to XYZ
    float x = _r * 0.412453f + _g * 0.357580f + _b * 0.180423f;
    float y = _r * 0.212671f + _g * 0.715160f + _b * 0.072169f;
    float z = _r * 0.019334f + _g * 0.119193f + _b * 0.950227f;

    x /= 0.950456f;
    float y3 = powf(y, 1.0f / 3.0f);
    z /= 1.088754f;

    // Improved condition checks for L*a*b conversion
    x = x > 0.008856f ? powf(x, 1.0f / 3.0f) : (7.787f * x + 0.13793f);
    y = y > 0.008856f ? y3 : 7.787f * y + 0.13793f;
    z = z > 0.008856f ? powf(z, 1.0f / 3.0f) : (7.787f * z + 0.13793f);

    // Calculate Lab values
    float l = y > 0.008856f ? (116.0f * y3 - 16.0f) : 903.3f * y;
    float a = (x - y) * 500.0f;
    float b = (y - z) * 200.0f;

    // Store the result in the output image
    float4 fPixel;
    fPixel.x = l;
    fPixel.y = a;
    fPixel.z = b;

    outputImg[offset] = fPixel;
}