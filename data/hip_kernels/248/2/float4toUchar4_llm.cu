#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void float4toUchar4(float4 *inputImage, uchar4 *outputImage, int width, int height)
{
    // Calculate 1D offset for global memory access.
    int offset = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    
    // Ensure offset is within image bounds
    if (offset < width * height) {
        float4 pixelf = inputImage[offset];
        uchar4 pixel;
        pixel.x = (unsigned char) pixelf.x; 
        pixel.y = (unsigned char) pixelf.y;
        pixel.z = (unsigned char) pixelf.z; 
        pixel.w = (unsigned char) pixelf.w;
        
        outputImage[offset] = pixel;
    }
}