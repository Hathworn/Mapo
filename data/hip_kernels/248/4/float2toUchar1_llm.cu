#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void float2toUchar1(float2 *inputImage, uchar1 *outputImage, int width, int height, int index)
{
    // Optimize offset calculation using 2D block and thread indexing
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
  
    // Check boundaries to avoid accessing out of bounds
    if (x < width && y < height) {
        int offset = y * width + x; // Simplified offset calculation for 2D grid

        float2 pixelf = inputImage[offset];
        float pixelfIndexed = (index == 0) ? pixelf.x : pixelf.y;

        uchar1 pixel;
        pixel.x = (unsigned char)pixelfIndexed;

        outputImage[offset] = pixel;
    }
}