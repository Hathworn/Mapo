#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void float1toUchar4(float1 *inputImage, uchar4 *outputImage, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = y * width + x;

    if (x < width && y < height) {  // Ensure within bounds
        float1 pixelf = inputImage[offset];
        unsigned char pixelValue = (unsigned char)pixelf.x;
        uchar4 pixel = {pixelValue, pixelValue, pixelValue, pixelValue};
        outputImage[offset] = pixel;
    }
}