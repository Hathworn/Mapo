#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rotate_3D(float* coords, size_t dim_z, size_t dim_y, size_t dim_x, float* rot_matrix) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = dim_x * dim_y * dim_z;

    if (index < total) {
        float old_z = coords[index];
        float old_y = coords[index + total];
        float old_x = coords[index + 2 * total];
        
        // Compute new coordinates using rotation matrix
        float new_z = old_z * rot_matrix[0] + old_y * rot_matrix[3] + old_x * rot_matrix[6];
        float new_y = old_z * rot_matrix[1] + old_y * rot_matrix[4] + old_x * rot_matrix[7];
        float new_x = old_z * rot_matrix[2] + old_y * rot_matrix[5] + old_x * rot_matrix[8];
        
        // Store the result back without using unnecessary __syncthreads
        coords[index] = new_z;
        coords[index + total] = new_y;
        coords[index + 2 * total] = new_x;
    }
}