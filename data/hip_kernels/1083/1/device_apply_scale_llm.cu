#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void device_apply_scale(float* __restrict__ coords, float scale, size_t total_size) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < total_size) {
        coords[i] *= scale; // Simplified in-place multiplication
    }
}