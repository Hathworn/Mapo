#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256

__device__ __forceinline__ int mirror(int index, int len){
    int s2 = 2 * len - 2;
    if(index < 0){
        index = s2 * (-index / s2) + index;
        return index <= 1 - len ? index + s2 : -index;
    }
    if(index >= len){
        index -= s2 * (index / s2);
        if(index >= len)
            index = s2 - index;
        return index;
    }
    return index;
}

__global__ void gussain_filter_z(float* random, float* kernel, int lw, size_t dim_z, size_t dim_y, size_t dim_x, int mode, float cval){
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = dim_x * dim_y * dim_z;

    if(index >= total) return; // Early exit for threads beyond total size

    size_t total_xy = dim_x * dim_y;
    size_t id_x = index % dim_x;
    size_t id_y = (index / dim_x) % dim_y;
    size_t id_z = (index / total_xy) % dim_z;
    size_t id_block = index / total;

    float new_pixel = 0;

    for(int i = -lw; i <= lw; i++){
        int id = id_z + i;
        if(mode == 0){
            if(id < 0 || id >= dim_z)
                new_pixel += cval * kernel[i + lw];
            else
                new_pixel += kernel[i + lw] *
                random[id_block * total + id * total_xy + id_y * dim_x + id_x];
        } else {
            id = mirror(id, dim_z);
            new_pixel += kernel[i + lw] *
            random[id_block * total + id * total_xy + id_y * dim_x + id_x];
        }
    }

    random[index] = new_pixel;
}