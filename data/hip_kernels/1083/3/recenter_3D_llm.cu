#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void recenter_3D(float* coords, size_t dim_z, size_t dim_y, size_t dim_x) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = dim_x * dim_y * dim_z;
    
    // Efficient memory access and avoiding re-computation
    if (index < total) {
        float half_dim_z = (float)dim_z / 2.0f;
        float half_dim_y = (float)dim_y / 2.0f;
        float half_dim_x = (float)dim_x / 2.0f;

        coords[index] += half_dim_z;
        coords[index + total] += half_dim_y;
        coords[index + 2 * total] += half_dim_x;
    }
    // Remove unnecessary __syncthreads(); as threads do not communicate
}