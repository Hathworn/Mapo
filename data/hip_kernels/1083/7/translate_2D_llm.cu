#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void translate_2D(float* coords, size_t dim_y, size_t dim_x, float seg_y, float seg_x) {
    // Compute the flattened index (2D index to 1D index)
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = dim_x * dim_y;
    
    // Ensure within bounds
    if(index < total) {
        // Perform translation by adding segment offsets
        coords[index] += seg_y;
        coords[index + total] += seg_x;
    }
    // __syncthreads() is unnecessary outside of conditional logic affecting memory access
}