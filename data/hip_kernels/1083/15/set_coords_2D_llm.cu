#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_coords_2D(float* coords, size_t y, size_t x) {
    // Calculate global thread index
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only proceed for valid index
    if (index < x * y) {
        size_t id_x = index % x;
        size_t id_y = index / x;

        // Assign computed coordinates
        size_t base_index = id_x + id_y * x;
        coords[base_index] = id_y - (float)y / 2.0;
        coords[base_index + x * y] = id_x - (float)x / 2.0; 
    }
    // Remove unnecessary __syncthreads() since no shared memory is used
}