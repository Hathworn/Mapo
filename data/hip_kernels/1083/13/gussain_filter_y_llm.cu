#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ int mirror(int index, int len) {
    int s2 = 2 * len - 2;
    if (index < 0) {
        index = s2 * (-index / s2) + index;
        return index <= 1 - len ? index + s2 : -index;
    }
    if (index >= len) {
        index -= s2 * (index / s2);
        if (index >= len)
            index = s2 - index;
        return index;
    }
    return index;
}

__global__ void gussain_filter_y(float* random, float* kernel, int lw, size_t dim_z, size_t dim_y, size_t dim_x, int mode, float cval) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = dim_x * dim_y * dim_z;
    size_t total_xy = dim_x * dim_y;

    // Early exit if index is out of bounds
    if (index >= total) return;

    size_t id_x = index % dim_x;
    size_t id_y = (index / dim_x) % dim_y;
    size_t id_z = (index / total_xy) % dim_z;
    size_t id_block = index / total;

    float new_pixel = 0.0f;

    // Optimize: Flatten the id calculation and loop only up to required range
    for (int i = -lw; i <= lw; ++i) {
        int id = id_y + i;
        float kernel_value = kernel[i + lw];

        if (mode == 0) {
            // Directly handle out of bounds situation
            if (id < 0 || id >= dim_y) {
                new_pixel += cval * kernel_value;
            } else {
                new_pixel += kernel_value *
                             random[id_block * total + id_z * total_xy + id * dim_x + id_x];
            }
        } else {
            // Use mirror boundary
            id = mirror(id, dim_y);
            new_pixel += kernel_value *
                         random[id_block * total + id_z * total_xy + id * dim_x + id_x];
        }
    }

    __syncthreads();
    random[index] = new_pixel;
}