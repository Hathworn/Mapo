#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_coords_3D(float* coords, size_t z, size_t y, size_t x) {
    // Calculate global thread index
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the index is within bounds
    if (index < x * y * z) {
        size_t id_x = index % x;
        size_t id_y = (index / x) % y;
        size_t id_z = index / (x * y);

        // Compute and assign the coordinates
        float zOffset = (float)z / 2.0;
        float yOffset = (float)y / 2.0;
        float xOffset = (float)x / 2.0;
        
        coords[index] = id_z - zOffset;
        coords[index + x * y * z] = id_y - yOffset;
        coords[index + 2 * x * y * z] = id_x - xOffset;
    }
    // Removing unnecessary __syncthreads()
}