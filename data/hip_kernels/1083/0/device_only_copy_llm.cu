#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_only_copy(float* output, float* input, size_t total_size) {
    // Calculate the global thread index
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop unrolling
    size_t stride = blockDim.x * gridDim.x;
    while (i < total_size) {
        output[i] = input[i];
        i += stride;
    }

    // Remove __syncthreads() as it's unnecessary for this task
}