#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ int mirror(int index, int len) {
    int s2 = 2 * len - 2;
    if (index < 0) {
        index = s2 * (-index / s2) + index;
        return index <= 1 - len ? index + s2 : -index;
    }
    if (index >= len) {
        index -= s2 * (index / s2);
        if (index >= len)
            index = s2 - index;
        return index;
    }
    return index;
}

__global__ void gussain_filter_x(float* random, float* kernel, int lw, size_t dim_z, size_t dim_y, size_t dim_x, int mode, float cval) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = dim_x * dim_y * dim_z;
    if (index >= total) return;  // Early exit for invalid threads

    size_t total_xy = dim_x * dim_y;
    size_t id_x = index % dim_x;
    size_t id_y = (index / dim_x) % dim_y;
    size_t id_z = index / total_xy;
    size_t id_block = index / total;
    float new_pixel = 0.0f;

    for (int i = -lw; i <= lw; ++i) {
        int id = id_x + i;
        if (mode == 0) {
            if (id < 0 || id >= dim_x)
                new_pixel += cval * kernel[i + lw];
            else
                new_pixel += kernel[i + lw] * random[id_block * total + id_z * total_xy + id_y * dim_x + id];
        } else {
            id = mirror(id, dim_x);
            new_pixel += kernel[i + lw] * random[id_block * total + id_z * total_xy + id_y * dim_x + id];
        }
    }
    // Writes the new pixel value
    random[index] = new_pixel;
}