#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void recenter_2D(float* coords, size_t dim_y, size_t dim_x) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within the bounds
    if (index < dim_x * dim_y) {
        coords[index] += dim_y * 0.5f;                       // Simplified division
        coords[index + dim_x * dim_y] += dim_x * 0.5f;       // Simplified division
    }
}