#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rotate_2D(float* coords, size_t dim_y, size_t dim_x, float cos_angle, float sin_angle) {
    // Calculate linear index
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = dim_x * dim_y;

    // Ensure index is within bounds
    if (index < total) {
        float old_y = coords[index];
        float old_x = coords[index + total];

        // Compute new coordinates
        float new_y = cos_angle * old_y + sin_angle * old_x;
        float new_x = -sin_angle * old_y + cos_angle * old_x;

        // Assign new coordinates
        coords[index] = new_y;
        coords[index + total] = new_x;
    }
}