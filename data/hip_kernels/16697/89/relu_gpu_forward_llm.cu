#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global thread index
    int stride = blockDim.x * gridDim.x; // Calculate the stride
    for (int i = tid; i < N; i += stride) { // Loop with a stride to cover all elements
        out[i] = in[i] > 0 ? in[i] : 0; // Apply ReLU operation
    }
}