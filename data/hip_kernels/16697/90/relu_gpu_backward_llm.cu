#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    // Utilize shared memory for potential performance improvement
    __shared__ float shared_outgrad[256];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Load outgrad into shared memory (assuming blockDim.x is 256)
    if (threadIdx.x < 256)
        shared_outgrad[threadIdx.x] = outgrad[tid];

    __syncthreads();

    // Only access shared memory within valid indices
    if (tid < N)
        ingrad[tid] = indata[tid] > 0 ? shared_outgrad[threadIdx.x] : 0;
}