#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fillUnusedDiagonal(float* Rmatrices, uint32_t numSamples, uint32_t subArraySize, const uint32_t* subArraySizes)
{
    int tIdx = threadIdx.x + blockDim.x * threadIdx.y; // Unique thread index within the block
    int sampleIdx = blockIdx.x + gridDim.x * blockIdx.y; // Adjusted to flatten blockIdx

    if (sampleIdx < numSamples)
    {
        int subArraySizeLocal = subArraySizes[sampleIdx];
        if (subArraySize > subArraySizeLocal)
        {
            int numelRfull = subArraySize * subArraySize;
            float* R = &Rmatrices[sampleIdx * numelRfull];
            float diagEntry = R[numelRfull - 1]; // Fetch once outside loop

            for (int diagIdx = subArraySizeLocal + tIdx; diagIdx < subArraySize; diagIdx += blockDim.x * blockDim.y)
            {
                int matrixIdx = diagIdx * (subArraySize + 1); // Compute flattened index for diagonal
                R[matrixIdx] = diagEntry; // Set unused diagonal entries
            }
        }
    }
}