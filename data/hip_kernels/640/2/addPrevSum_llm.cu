#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addPrevSum(unsigned int* blkSumsScan, unsigned int* blkScans, unsigned int n)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    
    // Iterate over threads to process multiple elements
    while (i < n)
    {
        blkScans[i] += blkSumsScan[blockIdx.x];
        i += blockDim.x * gridDim.x; // Advance by total number of threads
    }
}