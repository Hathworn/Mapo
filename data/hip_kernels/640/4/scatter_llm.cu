#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scatter(unsigned int *in, unsigned int *in_pos, unsigned int *out, unsigned int *out_pos, unsigned int n, unsigned int *d_histScan, unsigned int mask, unsigned int current_bits, unsigned int nBins) {
    extern __shared__ unsigned int min_Idx[];

    // Initialize min_Idx using a single loop to improve performance.
    for (int j = threadIdx.x; j < nBins; j += blockDim.x)
        min_Idx[j] = n;
    __syncthreads();

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        unsigned int bin = (in[i] >> current_bits) & mask;
        atomicMin(&min_Idx[bin], i);
    }
    __syncthreads();

    // Combine the two if conditions for better warp execution.
    if (i < n) {
        unsigned int bin = (in[i] >> current_bits) & mask;
        unsigned int index = d_histScan[blockIdx.x + bin * gridDim.x] + i - min_Idx[bin];
        
        // Ensure coalesced global memory access for outputs.
        out[index] = in[i];
        out_pos[index] = in_pos[i];
    }
}