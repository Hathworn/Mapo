#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void swap(unsigned int *in, unsigned int *in_pos, unsigned int *out, unsigned int *out_pos, unsigned int n)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce global memory accesses
    extern __shared__ unsigned int shared_mem[];
    
    if (i < n)
    {
        // Load inputs into shared memory
        shared_mem[threadIdx.x] = in[i];
        shared_mem[threadIdx.x + blockDim.x] = out[i];

        shared_mem[threadIdx.x + 2 * blockDim.x] = in_pos[i];
        shared_mem[threadIdx.x + 3 * blockDim.x] = out_pos[i];
        
        __syncthreads();

        // Perform swap using shared memory
        unsigned int temp_in = shared_mem[threadIdx.x] ^ shared_mem[threadIdx.x + blockDim.x];
        shared_mem[threadIdx.x] = temp_in ^ shared_mem[threadIdx.x];
        shared_mem[threadIdx.x + blockDim.x] = temp_in;

        unsigned int temp_in_pos = shared_mem[threadIdx.x + 2 * blockDim.x] ^ shared_mem[threadIdx.x + 3 * blockDim.x];
        shared_mem[threadIdx.x + 2 * blockDim.x] = temp_in_pos ^ shared_mem[threadIdx.x + 2 * blockDim.x];
        shared_mem[threadIdx.x + 3 * blockDim.x] = temp_in_pos;

        __syncthreads();

        // Save outputs back to global memory
        in[i] = shared_mem[threadIdx.x];
        out[i] = shared_mem[threadIdx.x + blockDim.x];

        in_pos[i] = shared_mem[threadIdx.x + 2 * blockDim.x];
        out_pos[i] = shared_mem[threadIdx.x + 3 * blockDim.x];
    }
}