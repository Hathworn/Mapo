#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogram(unsigned int* in, unsigned int* hist, int n, unsigned int nBins, unsigned int mask, unsigned int current_bits)
{
    extern __shared__ unsigned int s_local_hist[];

    // Initialize shared memory with zeros
    for (int j = threadIdx.x; j < nBins; j += blockDim.x)
    {
        s_local_hist[j] = 0;
    }

    __syncthreads();

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        unsigned int bin = (in[i] >> current_bits) & mask;
        // Atomic addition in shared memory
        atomicAdd(&s_local_hist[bin], 1);
    }
    __syncthreads();

    // Write shared memory histogram to global memory
    for (int bin = threadIdx.x; bin < nBins; bin += blockDim.x)
    {
        hist[bin * gridDim.x + blockIdx.x] = s_local_hist[bin];
    }
}