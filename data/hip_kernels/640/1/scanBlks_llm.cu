#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scanBlks(unsigned int *in, unsigned int *out, unsigned int n, unsigned int *blkSums)
{
    extern __shared__ int blkData[];
    int i1 = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    int i2 = i1 + blockDim.x;

    // Load input into shared memory
    if (i1 < n) blkData[threadIdx.x] = in[i1];
    else blkData[threadIdx.x] = 0; // Handle boundary

    if (i2 < n) blkData[threadIdx.x + blockDim.x] = in[i2];
    else blkData[threadIdx.x + blockDim.x] = 0; // Handle boundary

    __syncthreads();

    // Up-sweep phase
    for (int stride = 1; stride < 2 * blockDim.x; stride *= 2) {
        int index = (threadIdx.x + 1) * 2 * stride - 1;

        if (index < 2 * blockDim.x) 
            blkData[index] += blkData[index - stride];

        __syncthreads();
    }

    // Down-sweep phase
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        int index = (threadIdx.x + 1) * 2 * stride - 1 + stride;

        if (index < 2 * blockDim.x) 
            blkData[index] += blkData[index - stride];

        __syncthreads();
    }

    // Write results to global memory
    if (i1 < n) out[i1] = blkData[threadIdx.x];
    if (i2 < n) out[i2] = blkData[threadIdx.x + blockDim.x];

    // Store block sums for further processing
    if (blkSums != NULL && threadIdx.x == 0) 
        blkSums[blockIdx.x] = blkData[2 * blockDim.x - 1];
}