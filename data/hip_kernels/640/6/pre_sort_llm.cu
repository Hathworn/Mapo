#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pre_sort(unsigned int *in, unsigned int *in_pos, unsigned int *out, unsigned int *out_pos, unsigned int n, unsigned int nBins, unsigned int mask, unsigned int current_bits, unsigned int *d_hist)
{
    extern __shared__ unsigned int pre_sort_blk_data[];
    unsigned int* blk_value = pre_sort_blk_data;
    unsigned int* blk_pos = pre_sort_blk_data + blockDim.x;
    unsigned int* blk_hist = pre_sort_blk_data + 2 * blockDim.x;
    unsigned int* blk_Scan = pre_sort_blk_data + nBins + 2 * blockDim.x;

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        blk_value[threadIdx.x] = in[i];
        blk_pos[threadIdx.x] = in_pos[i];
    }
    __syncthreads();

    // Initialize histogram
    for (int j = threadIdx.x; j < nBins; j += blockDim.x)
    {
        blk_hist[j] = 0;
        blk_Scan[j] = 0;
    }
    __syncthreads();

    if (i < n)
    {
        unsigned int bin = (blk_value[threadIdx.x] >> current_bits) & mask;
        atomicAdd(&blk_hist[bin], 1);
    }
    __syncthreads();

    // Exclusive scan for histogram
    for (int stride = 1; stride < nBins; stride *= 2)
    {
        unsigned int temp = 0;
        int j = threadIdx.x + stride;
        if (j < nBins) temp = blk_Scan[j - stride];
        __syncthreads();
        if (j < nBins) blk_Scan[j] += temp;
        __syncthreads();
    }

    // Adjust scan results
    if (threadIdx.x < nBins)
        blk_Scan[threadIdx.x] = (threadIdx.x == 0) ? 0 : blk_Scan[threadIdx.x - 1];
    __syncthreads();

    if (i < n)
    {
        unsigned int bin = (blk_value[threadIdx.x] >> current_bits) & mask;
        unsigned int pos = atomicAdd(&blk_Scan[bin], 1);
        out[pos + blockIdx.x * blockDim.x] = blk_value[threadIdx.x];
        out_pos[pos + blockIdx.x * blockDim.x] = blk_pos[threadIdx.x];
    }
}