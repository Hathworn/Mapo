#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void exclusive_scan(unsigned int *in, unsigned int *out, int n)
{
    // Use shared memory for increased access speed
    __shared__ unsigned int temp[256]; // Assume blockDim.x <= 256
    
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < n)
    {
        // Load input into shared memory
        temp[threadIdx.x] = in[i];
        
        __syncthreads(); // Ensure all threads have written
        
        // Perform exclusive scan in shared memory
        unsigned int offset = 1;
        while (offset < blockDim.x)
        {
            if (threadIdx.x >= offset)
            {
                temp[threadIdx.x] += temp[threadIdx.x - offset];
            }
            __syncthreads(); // Synchronize after each iteration
            
            offset <<= 1; // Double the offset
        }
        
        if (threadIdx.x == 0)
        {
            temp[0] = 0; // Ensure first element is zero for exclusive scan
        }
        
        __syncthreads(); // Final synchronization
        
        out[i] = temp[threadIdx.x]; // Write scan result to global memory
    }
}