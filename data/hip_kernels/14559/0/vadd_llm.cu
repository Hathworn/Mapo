#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// a simple HIP kernel to add two vectors
extern "C"
{

} // extern "C"

// Optimized kernel: Utilize shared memory and check bounds
__global__ void vadd(const float *a, const float *b, float *c, int n)
{
    extern __shared__ float shared_mem[];
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (i < n) {
        shared_mem[threadIdx.x] = a[i] + b[i]; // Load into shared memory
        __syncthreads(); // Ensure all threads have completed writing to shared memory
        c[i] = shared_mem[threadIdx.x]; // Write back to global memory
    }
}