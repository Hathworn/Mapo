#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prefix_sum_scan(uint* dev_main_array, uint* dev_auxiliary_array, const uint array_size)
{
    // Calculate the global index
    uint element = blockIdx.x * blockDim.x + threadIdx.x;

    // Correct offset for processing the second block onwards
    element += blockDim.x;

    // Ensure within bounds
    if (element < array_size) {
        uint cluster_offset = dev_auxiliary_array[blockIdx.x + 1];
        dev_main_array[element] += cluster_offset;
    }
}