#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void down_sweep_512(uint* data_block)
{
    for (uint i = 512; i >= 2; i >>= 1) {
        // Minimize memory accesses
        uint j_stride = blockDim.x * (i >> 1);
        for (uint j = 0; j < (511 + blockDim.x) / i; ++j) {
            const auto element = 511 - (j * blockDim.x + threadIdx.x) * i;
            if (element < 512) {
                const auto other_element = element - (i >> 1);
                const auto value = data_block[other_element];
                data_block[other_element] = data_block[element];
                data_block[element] += value;
            }
        }
        __syncthreads();
    }
}

__device__ void up_sweep_512(uint* data_block)
{
    uint starting_elem = 1;
    for (uint i = 2; i <= 512; i <<= 1) {
        // Reduce index calculations
        uint j_stride = blockDim.x * (i >> 1);
        for (uint j = 0; j < (511 + blockDim.x) / i; ++j) {
            const uint element = starting_elem + (j * blockDim.x + threadIdx.x) * i;
            if (element < 512) {
                data_block[element] += data_block[element - (i >> 1)];
            }
        }
        starting_elem += i;
        __syncthreads();
    }
}

__global__ void prefix_sum_reduce(uint* dev_main_array, uint* dev_auxiliary_array, const uint array_size)
{
    // Use a data block size of 512
    __shared__ uint data_block[512];

    // Calculate thread offsets once
    uint thread_offset = threadIdx.x + blockDim.x;
    const uint last_block = array_size >> 9;
    if (blockIdx.x < last_block) {
        const uint first_elem = blockIdx.x << 9;

        // Load elements into shared memory, add prev_last_elem
        data_block[threadIdx.x] = dev_main_array[first_elem + threadIdx.x];
        data_block[thread_offset] = dev_main_array[first_elem + thread_offset];

        __syncthreads();

        up_sweep_512((uint*) &data_block[0]);

        if (threadIdx.x == 0) {
            dev_auxiliary_array[blockIdx.x] = data_block[511];
            data_block[511] = 0;
        }

        __syncthreads();

        down_sweep_512((uint*) &data_block[0]);

        // Store back elements
        dev_main_array[first_elem + threadIdx.x] = data_block[threadIdx.x];
        dev_main_array[first_elem + threadOffset] = data_block[thread_offset];

        __syncthreads();
    }
    // Last block is special because
    else {
        const auto elements_remaining = array_size & 0x1FF; // % 512
        if (elements_remaining > 0) {
            const auto first_elem = array_size - elements_remaining;

            // Initialize all elements to zero
            data_block[threadIdx.x] = 0;
            data_block[thread_offset] = 0;

            // Load elements
            const auto elem_index = first_elem + threadIdx.x;
            if (elem_index < array_size) {
                data_block[threadIdx.x] = dev_main_array[elem_index];
            }
            if ((elem_index + blockDim.x) < array_size) {
                data_block[thread_offset] = dev_main_array[elem_index + blockDim.x];
            }

            __syncthreads();

            up_sweep_512((uint*) &data_block[0]);

            // Store sum of all elements
            if (threadIdx.x == 0) {
                dev_auxiliary_array[blockIdx.x] = data_block[511];
                data_block[511] = 0;
            }

            __syncthreads();

            down_sweep_512((uint*) &data_block[0]);

            // Store back elements
            if (elem_index < array_size) {
                dev_main_array[elem_index] = data_block[threadIdx.x];
            }
            if ((elem_index + blockDim.x) < array_size) {
                dev_main_array[elem_index + blockDim.x] = data_block[thread_offset];
            }
        }
    }
}