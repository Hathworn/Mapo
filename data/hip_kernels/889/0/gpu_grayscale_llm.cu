#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_grayscale(int width, int height, float *image, float *image_out)
{
    // Calculate the linear index based on block and thread indices
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check that the index is within the bounds of the image
    if (idx < width * height) {
        // Each pixel has 3 floats for R, G, and B
        int pixel_start_idx = idx * 3;

        // Extract RGB values
        float r = image[pixel_start_idx];
        float g = image[pixel_start_idx + 1];
        float b = image[pixel_start_idx + 2];

        // Compute grayscale value using luminosity method
        image_out[idx] = 0.21f * r + 0.72f * g + 0.07f * b;
    }
}