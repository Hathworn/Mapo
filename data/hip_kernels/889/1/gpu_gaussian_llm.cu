#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float gpu_applyFilter(float *image, int stride, float *matrix, int filter_dim)
{
    // Optimize gpu_applyFilter by unrolling loops and removing extra calculations
    float sum = 0.0f;
    for (int y = 0; y < filter_dim; ++y) {
        for (int x = 0; x < filter_dim; ++x) {
            sum += image[y * stride + x] * matrix[y * filter_dim + x];
        }
    }
    return sum;
}

__global__ void gpu_gaussian(int width, int height, float *image, float *image_out)
{
    __shared__ float gaussian[9];  // Use shared memory for faster access
    if (threadIdx.x < 9) {
        gaussian[threadIdx.x] = 1.0f / 16.0f * (threadIdx.x % 2 + threadIdx.x / 2);  // Precompute filter
    }
    __syncthreads();

    int index_x = blockIdx.x * blockDim.x + threadIdx.x;
    int index_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (index_x < (width - 2) && index_y < (height - 2))
    {
        int offset_t = index_y * width + index_x;
        int offset = (index_y + 1) * width + (index_x + 1);

        image_out[offset] = gpu_applyFilter(&image[offset_t], width, gaussian, 3);
    }
}