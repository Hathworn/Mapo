#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_helloworld()
{
    // Cache thread index to a register for faster access
    int threadId = threadIdx.x;
    
    // Use more efficient printf
    printf("Hello from the GPU! My threadId is %d\n", threadId);
}