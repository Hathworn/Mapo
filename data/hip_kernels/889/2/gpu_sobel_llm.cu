#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_sobel(int width, int height, float *image, float *image_out)
{
    // Get the row and column from the block and thread indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within image bounds
    if (x >= 1 && x < width - 1 && y >= 1 && y < height - 1)
    {
        // Compute Sobel operators
        float Gx = 
            -1 * image[(y - 1) * width + (x - 1)] + 
             1 * image[(y - 1) * width + (x + 1)] + 
            -2 * image[(y    ) * width + (x - 1)] + 
             2 * image[(y    ) * width + (x + 1)] + 
            -1 * image[(y + 1) * width + (x - 1)] + 
             1 * image[(y + 1) * width + (x + 1)];

        float Gy = 
            -1 * image[(y - 1) * width + (x - 1)] + 
            -2 * image[(y - 1) * width + (x    )] + 
            -1 * image[(y - 1) * width + (x + 1)] + 
             1 * image[(y + 1) * width + (x - 1)] + 
             2 * image[(y + 1) * width + (x    )] + 
             1 * image[(y + 1) * width + (x + 1)];

        // Calculate magnitude of gradient
        image_out[y * width + x] = sqrt(Gx * Gx + Gy * Gy);
    }
}