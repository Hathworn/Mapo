#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void copy_kernel_frombuf(char *dest, char *src, int rx_s, int rx_e, int ry_s, int ry_e, int rz_s, int rz_e, int x_step, int y_step, int z_step, int size_x, int size_y, int size_z, int buf_strides_x, int buf_strides_y, int buf_strides_z, int type_size, int dim, int OPS_soa) {

    // Cache the block and thread indices
    int idx_z = rz_s + z_step * (blockDim.z * blockIdx.z + threadIdx.z);
    int idx_y = ry_s + y_step * (blockDim.y * blockIdx.y + threadIdx.y);
    int idx_x = rx_s + x_step * (blockDim.x * blockIdx.x + threadIdx.x);

    // Conditional boundary check using a single branch for each dimension
    if ((x_step == 1 ? (idx_x < rx_e) : (idx_x > rx_e)) &&
        (y_step == 1 ? (idx_y < ry_e) : (idx_y > ry_e)) &&
        (z_step == 1 ? (idx_z < rz_e) : (idx_z > rz_e))) {

        // Compute initial offset for dest
        size_t dest_offset = (idx_z * size_x * size_y + idx_y * size_x + idx_x) * type_size;
        if (!OPS_soa) dest_offset *= dim;
        dest += dest_offset;

        // Compute initial offset for src
        src += (((idx_z - rz_s) * z_step * buf_strides_z +
                 (idx_y - ry_s) * y_step * buf_strides_y +
                 (idx_x - rx_s) * x_step * buf_strides_x) * type_size * dim);

        // Unroll loop when dim is a known compile-time constant
        for (int d = 0; d < dim; d++) {
            memcpy(dest, src + d * type_size, type_size);
            // Efficient pointer arithmetic based on OPS_soa flag
            dest += (OPS_soa) ? (size_x * size_y * size_z * type_size) : type_size;
        }
    }
}