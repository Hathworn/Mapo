#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

int halo_buffer_size = 0;
char *halo_buffer_d = NULL;

// Optimized GPU kernel
__global__ void copy_kernel_tobuf(char *dest, char *src, int rx_s, int rx_e, int ry_s, int ry_e, int rz_s, int rz_e, int x_step, int y_step, int z_step, int size_x, int size_y, int size_z, int buf_strides_x, int buf_strides_y, int buf_strides_z, int type_size, int dim, int OPS_soa) {

    // Calculate global indices
    int idx_z = rz_s + z_step * (blockDim.z * blockIdx.z + threadIdx.z);
    int idx_y = ry_s + y_step * (blockDim.y * blockIdx.y + threadIdx.y);
    int idx_x = rx_s + x_step * (blockDim.x * blockIdx.x + threadIdx.x);

    // Check if within bounds
    if ((x_step == 1 ? idx_x < rx_e : idx_x > rx_e) &&
        (y_step == 1 ? idx_y < ry_e : idx_y > ry_e) &&
        (z_step == 1 ? idx_z < rz_e : idx_z > rz_e)) {

        // Compute source and destination pointers
        size_t offset = (idx_z * size_x * size_y + idx_y * size_x + idx_x) * type_size;
        if (OPS_soa) src += offset;
        else src += offset * dim;
        dest += ((idx_z - rz_s) * z_step * buf_strides_z +
                 (idx_y - ry_s) * y_step * buf_strides_y +
                 (idx_x - rx_s) * x_step * buf_strides_x) *
                type_size * dim;
        
        // Copy data using loop unrolling
        for (int d = 0; d < dim; d++) {
            memcpy(dest + d * type_size, src, type_size);
            if (OPS_soa) src += size_x * size_y * size_z * type_size;
            else src += type_size;
        }
    }
}