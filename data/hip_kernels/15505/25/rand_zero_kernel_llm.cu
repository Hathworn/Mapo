#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rand_zero_kernel(float *data, int n, float p, hiprandStatePhilox4_32_10_t *states) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;

    // Only perform operations if x is less than n to avoid unnecessary calculations
    if (x < n) {
        hiprandStatePhilox4_32_10_t &state = states[x];
        x *= 4;

        // Unroll loop manually for potential performance gain
        float4 vals = hiprand_uniform4(&state);
        if (reinterpret_cast<float*>(&vals)[0] < p && (x + 0) < n) data[x + 0] = 0;
        if (reinterpret_cast<float*>(&vals)[1] < p && (x + 1) < n) data[x + 1] = 0;
        if (reinterpret_cast<float*>(&vals)[2] < p && (x + 2) < n) data[x + 2] = 0;
        if (reinterpret_cast<float*>(&vals)[3] < p && (x + 3) < n) data[x + 3] = 0;
    }
}