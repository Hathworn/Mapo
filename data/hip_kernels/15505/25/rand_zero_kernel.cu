#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rand_zero_kernel(float *data, int n, float p, hiprandStatePhilox4_32_10_t *states) {
int x(threadIdx.x + blockDim.x * blockIdx.x);

hiprandStatePhilox4_32_10_t &state(states[x]);

x *= 4;
float4 vals = hiprand_uniform4(&state);
for (int i(0); i < 4; ++i, ++x) {
if (x >= n) return;
if (reinterpret_cast<float*>(&vals)[i] < p)
data[x] = 0;
}
}