#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void exp_kernel(float *v, int n) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary condition and apply exp calculation
    if (x < n) {
        v[x] = expf(v[x]); // Use expf for single-precision
    }
}