#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rand_init_kernel(int seed, hiprandStatePhilox4_32_10_t *states, int n) {
int x(threadIdx.x + blockDim.x * blockIdx.x);

if (x < n)
hiprand_init(seed, x, 0, &states[x]);
}