#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ void copy_c(float const *in, float *out, int slicesizein, int slicesizeout, int C) {
    // Optimize with loop unrolling
    #pragma unroll
    for (size_t c(0); c < C; ++c)
        out[c * slicesizeout] = in[c * slicesizein];
}

__device__ __forceinline__ void add_c(float const *in, float *out, int slicesizein, int slicesizeout, int C) {
    // Optimize with loop unrolling
    #pragma unroll
    for (size_t c(0); c < C; ++c)
        out[c * slicesizeout] += in[c * slicesizein];
}

__device__ __forceinline__ int get_index(int X, int Y, int Z, int C, int x, int y, int z) {
    return z * (C * X * Y) + y * X + x;
}

__global__ void unshift_kernel(float const *in, float *out, int X, int Y, int C, int dx, int dy, float const beta) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;       // Compute global index
    int x = idx % X;                                       // Compute x from idx
    int y = idx / X;                                       // Compute y from idx

    int x_to(x + dx), y_to(y + dy);

    if (x >= X || y >= Y || x_to >= X || y_to >= Y || x_to < 0 || y_to < 0)
        return;

    int in_index = get_index(X, Y, 1, C, x_to, y_to, 0);   // Precompute in index
    int out_index = get_index(X, Y, 1, C, x, y, 0);        // Precompute out index

    if (beta > 0)
        add_c(in + in_index, out + out_index, X * Y, X * Y, C);
    else
        copy_c(in + in_index, out + out_index, X * Y, X * Y, C);
}