#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void times_scalard(double *v, double other, int n) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    // Use a stride loop to process elements
    for (int i = x; i < n; i += blockDim.x * gridDim.x) {
        v[i] *= other;
    }
}