#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ void copy_c(float const *in, float *out, int slicesizein, int slicesizeout, int C) {
    // Optimize inner loop by using pragma unroll
    #pragma unroll
    for (size_t c = 0; c < C; ++c)
        out[c * slicesizeout] = in[c * slicesizein];
}

__device__ __forceinline__ void add_c(float const *in, float *out, int slicesizein, int slicesizeout, int C) {
    // Optimize inner loop by using pragma unroll
    #pragma unroll
    for (size_t c = 0; c < C; ++c)
        out[c * slicesizeout] += in[c * slicesizein];
}

__device__ __forceinline__ int get_index(int X, int Y, int Z, int C, int x, int y, int z) {
    return z * (C * X * Y) + y * X + x;
}

__global__ void shift_kernel(float const *in, float *out, int X, int Y, int C, int dx, int dy, float const beta) {
    // Optimize index calculation through shared memory
    extern __shared__ int sharedIdx[];
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int x = tid % X;
    int y = tid / X;

    if (x >= X || y >= Y) return;

    int x_to = x + dx;
    int y_to = y + dy;

    if (x_to < 0 || x_to >= X || y_to < 0 || y_to >= Y) return;

    float const *in_ptr = in + get_index(X, Y, 1, C, x, y, 0);
    float *out_ptr = out + get_index(X, Y, 1, C, x_to, y_to, 0);

    if (beta > 0) {
        add_c(in_ptr, out_ptr, X * Y, X * Y, C);
    } else {
        copy_c(in_ptr, out_ptr, X * Y, X * Y, C);
    }
}