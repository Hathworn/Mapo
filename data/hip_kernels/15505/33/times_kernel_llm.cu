#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void times_kernel(float *v, float *other, int n) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if (x < n) {
        v[x] *= other[x];
    }
}