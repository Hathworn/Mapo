#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void abs_kerneld(double *v, int n) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    if (x < n) {
        // Use ternary operator for branchless optimization
        v[x] = v[x] >= 0 ? v[x] : -v[x];
    }
}