#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void exp_kerneld(double *v, int n) {
    // Calculate global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a loop to allow each thread to process multiple elements
    while (x < n) {
        v[x] = exp(v[x]);
        x += blockDim.x * gridDim.x; // Move to the next set of elements
    }
}