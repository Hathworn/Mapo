#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divide_kerneld(double *v, double *other, int n) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate thread index
    if (x < n) {  // Only proceed if within bounds
        v[x] /= other[x];  // Element-wise division
    }
}