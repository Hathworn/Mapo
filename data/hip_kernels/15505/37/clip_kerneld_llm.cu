#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clip_kerneld(double *v, int n, double limit) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;  // Corrected initialization syntax
    if (x < n) {  // Inverted condition for better thread performance
        double val = v[x];
        v[x] = (val > limit) ? limit : ((val < -limit) ? -limit : val);  // Store v[x] in a register
    }
}