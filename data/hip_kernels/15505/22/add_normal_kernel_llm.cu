#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_normal_kernel(int seed, float *data, int n, float mean, float std) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x; // Calculate global thread index
    if (idx >= n) return; // Exit if thread index is out of data range

    hiprandState state;
    hiprand_init(seed, idx, 0, &state); // Use unique seed per thread

    // Each thread processes one element based on its index
    data[idx] += hiprand_normal(&state) * std + mean;
}
```
