#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normal_kernel(int seed, float *data, int n, float mean, float std) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if thread index is within bounds
    if (id >= n) return;
    
    hiprandState state;
    // Initialize state with thread-specific seed for parallel generation
    hiprand_init(seed, id, 0, &state);
    
    // Each thread generates one random normal value
    data[id] = hiprand_normal(&state) * std + mean;
}