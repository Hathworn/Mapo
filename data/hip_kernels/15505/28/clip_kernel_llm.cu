#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void clip_kernel(float *v, int n, float limit) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Return early if index is out of bounds
    if (x >= n) return;

    float val = v[x];
    
    // Simplify condition for clarity and potential optimization
    if (val > limit) {
        v[x] = limit;
    } else if (val < -limit) {
        v[x] = -limit;
    }
}