#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sqrt_kerneld(double *v, int n) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < n) {
        // Calculate square root within bounds
        v[x] = sqrt(v[x]);
    }
}