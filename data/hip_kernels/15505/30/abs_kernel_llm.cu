#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void abs_kernel(float *v, int n) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure valid index and use fast inline function for abs
    if (x < n) {
        v[x] = fabsf(v[x]);
    }
}