#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void times_scalar(float *v, float other, int n) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory to reduce global memory accesses
    __shared__ float tile[256];
    if (x < n) {
        tile[threadIdx.x] = v[x] * other;
    }
    __syncthreads();

    if (x < n) {
        v[x] = tile[threadIdx.x];
    }
}