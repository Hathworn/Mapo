#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void times_kerneld(double * __restrict__ v, const double * __restrict__ other, int n) {
    // Calculate global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary condition and perform multiplication
    if (x < n) {
        v[x] *= other[x];
    }
}