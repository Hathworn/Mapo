#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_normal_kerneld(int seed, double *data, int n, double mean, double std) {
    // Calculate global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize state for each thread
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    
    // Use thread ID to access array elements directly
    for (size_t i = idx; i < n; i += gridDim.x * blockDim.x) {
        data[i] += hiprand_normal_double(&state) * std + mean;
    }
}