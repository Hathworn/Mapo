#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void divide_kernel(float *v, float *other, int n) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Use warp-level operations for improved parallelism
    if (x < n) {
        v[x] /= other[x];
    }
}