#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_scalar(float *v, float other, int n) {
    // Calculate global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid stride loop for more parallelism
    for (int idx = x; idx < n; idx += blockDim.x * gridDim.x) {
        v[idx] += other;
    }
}