#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(float *v, int n, float e) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    if (x < n) {
        // Use fast math intrinsic for power operation if applicable
        v[x] = powf(v[x], e);
    }
}