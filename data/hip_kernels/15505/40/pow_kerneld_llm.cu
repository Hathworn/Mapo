#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kerneld(double *v, int n, double e) {
    int x = threadIdx.x + blockDim.x * blockIdx.x;
    if (x < n) {
        // Use fast intrinsic for power calculation
        v[x] = __dpow(v[x], e);
    }
}