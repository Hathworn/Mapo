#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normal_kerneld(int seed, double *data, int n, double mean, double std) {
    // Each thread computes its own random number, improving parallelism
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= n) return;

    // Use grid-stride loop to handle larger data sizes
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);
    for (size_t i = idx; i < n; i += blockDim.x * gridDim.x) {
        data[i] = hiprand_normal_double(&state) * std + mean;
    }
}