#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normal_kerneld(int seed, double *data, int n, double mean, double std) {
if (threadIdx.x != 0) return;
hiprandState state;
hiprand_init(seed, 0, 0, &state);
for (size_t i(0); i < n; ++i)
data[i] = hiprand_normal_double(&state) * std + mean;
}