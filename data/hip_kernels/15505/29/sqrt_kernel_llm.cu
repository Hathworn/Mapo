#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sqrt_kernel(float *v, int n) {
    // Efficient calculation of thread index x 
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if x is within bounds
    if (x < n) {
        // Calculate square root of v[x]
        v[x] = sqrtf(v[x]);
    }
}