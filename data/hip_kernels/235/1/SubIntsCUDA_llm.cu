#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;
# define size 5

// Optimized global function
__global__ void SubIntsCUDA(int *a, int *b)
{
    // Optimize the calculation of the thread index using auto GPU variable
    int tid = hipThreadIdx_x + hipBlockIdx_x * hipBlockDim_x;

    // Add boundary check to prevent out-of-bounds memory access
    if (tid < size) 
    {
        // Perform the subtraction operation
        b[tid] = a[tid] - b[tid];
    }
}