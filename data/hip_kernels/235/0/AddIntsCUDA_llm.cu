#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
# define size 5

// Global functions

//********************************************************
__global__ void AddIntsCUDA(int *a, int *b)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use if condition to prevent out of bounds
    if (tid < size) {
        a[tid] += b[tid];
    }
}