#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel by ensuring thread index calculation and use of shared memory
__global__ void double_value(double *x, double *y)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Consider using shared memory for further optimization if data reused
    y[idx] = 2.0 * x[idx];
}