#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//设备端代码
__global__ void cvlUnit(const char *imgR,const char *imgG,const char *imgB,const char *core, char *outR,char *outG,char *outB,int lenX,int lenY,int lenCore)
{
    // Calculate coordinates
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int index = y * lenX + x;
    
    // Return if out of bounds
    if (x >= lenX || y >= lenY) return;

    int halfCore = lenCore / 2;
    if (x < halfCore || x >= lenX - halfCore || y < halfCore || y >= lenY - halfCore) {
        outR[index] = imgR[index];
        outG[index] = imgG[index];
        outB[index] = imgB[index];
        return;
    }
    
    int sumR = 0, sumG = 0, sumB = 0;
    
    // Loop unrolling for kernel calculations
    for (int i = 0; i < lenCore; ++i) {
        for (int j = 0; j < lenCore; ++j) {
            int tmpX = x + i - halfCore;
            int tmpY = y + j - halfCore;
            int coreValue = core[j * lenCore + i];
            int idx = tmpY * lenX + tmpX;
            
            sumR += imgR[idx] * coreValue;
            sumG += imgG[idx] * coreValue;
            sumB += imgB[idx] * coreValue;
        }
    }

    int coreArea = lenCore * lenCore;
    outR[index] = (char)(sumR / coreArea);
    outG[index] = (char)(sumG / coreArea);
    outB[index] = (char)(sumB / coreArea);
}