#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cvlUnit(const char *imgR,const char *imgG,const char *imgB,const char *core, char *outR,char *outG,char *outB,int lenX,int lenY,int lenCore)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int index = y * lenX + x;

    // Return if out of image bounds
    if (x >= lenX || y >= lenY) return;

    // Return original pixel if out of convolution bounds
    if (x - lenCore / 2 < 0 || x + lenCore / 2 >= lenX || y - lenCore / 2 < 0 || y + lenCore / 2 >= lenY) {
        outR[index] = imgR[index];
        outG[index] = imgG[index];
        outB[index] = imgB[index];
        return;
    }

    int sumR = 0, sumG = 0, sumB = 0;

    // Loop unrolling for core operations
    for (int i = 0; i < lenCore; i++) {
        for (int j = 0; j < lenCore; j++) {
            int tmpX = x - lenCore / 2 + i;
            int tmpY = y - lenCore / 2 + j;
            int coreVal = core[j * lenCore + i];

            // Accumulating sums for R, G, B channels
            sumR += imgR[tmpY * lenX + tmpX] * coreVal;
            sumG += imgG[tmpY * lenX + tmpX] * coreVal;
            sumB += imgB[tmpY * lenX + tmpX] * coreVal;
        }
    }

    // Normalize and assign to output
    int normFactor = lenCore * lenCore;
    outR[index] = (char)(sumR / normFactor);
    outG[index] = (char)(sumG / normFactor);
    outB[index] = (char)(sumB / normFactor);
}