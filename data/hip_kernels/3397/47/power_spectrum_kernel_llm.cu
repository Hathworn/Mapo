#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    // Use shared memory for better memory access pattern
    __shared__ float2 shared_data[CU1DBLOCK];

    for (int idx = thread_id; idx < half_length; idx += CU1DBLOCK) {
        // Ignore special case with proper read index
        if (idx == 0) continue;

        shared_data[thread_id] = reinterpret_cast<float2 *>(Ar)[idx];
        __syncthreads();  // Ensure all threads have loaded their data

        float2 val = shared_data[thread_id];
        float ret = val.x * val.x + val.y * val.y;
        if (use_power) {
            Aw[idx] = ret;
        } else {
            Aw[idx] = sqrtf(ret);
        }
        __syncthreads(); // Ensure all threads have finished computation
    }

    // Handle special case with correct index
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabs(real);
            Aw[half_length] = fabs(im);
        }
    }
}