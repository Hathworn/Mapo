#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    const float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    // Use shared memory for improved memory access efficiency
    extern __shared__ float2 shared_mem[];
    
    // Load data into shared memory
    for(int i = thread_id; i < half_length; i += blockDim.x) {
        shared_mem[i] = reinterpret_cast<const float2 *>(Ar)[i];
    }
    __syncthreads();

    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        // Ignore special case
        if (idx == 0) continue;

        // Compute power spectrum
        float2 val = shared_mem[idx];
        float ret = val.x * val.x + val.y * val.y;
        if (use_power) {
            Aw[idx] = ret;
        } else {
            Aw[idx] = sqrtf(ret);
        }
    }

    // Handle special case
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabs(real);
            Aw[half_length] = fabs(im);
        }
    }
}