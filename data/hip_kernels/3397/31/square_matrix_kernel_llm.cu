#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Calculate row and column index based on block and thread indices
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check within bounds before processing
    if (i < num_rows && j < num_cols) {
        float f = feats[i * ldf + j];
        feats_sq[i * lds + j] = f * f; // Store the square of the element
    }
}