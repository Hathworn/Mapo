#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    float cur_weight = *cur_tot_weight;
    float new_num_frames = old_num_frames + cur_weight;
    float prior_scale_change = 1.0f;

    if (max_count != 0) {
        float old_prior_scale = max(old_num_frames, (float)max_count) / max_count;
        float new_prior_scale = max(new_num_frames, (float)max_count) / max_count;
        prior_scale_change += new_prior_scale - old_prior_scale;
    }

    int32_t threadId = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate thread ID in grid
    int32_t stride = blockDim.x * gridDim.x;                   // Calculate stride across grid

    // Optimize: Use stride loop for coalesced memory access
    for (int32_t i = threadId; i < n; i += stride) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        quadratic[diag_idx] += prior_scale_change;
    }

    // Optimize: Use atomicAdd for concurrent access
    if (threadId == 0) {
        atomicAdd(&linear[0], prior_offset * prior_scale_change);
    }
}