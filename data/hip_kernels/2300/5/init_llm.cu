#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(uint32_t *v, uint32_t val, uint32_t n) {
    // Calculate global thread index
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check boundary condition out of loop for increased efficiency
    if (i < n) {
        v[i] = val;
    }
    // Use a single thread to print, reducing unnecessary checks
    if (i == 0) {
        printf("init\n");
    }
}