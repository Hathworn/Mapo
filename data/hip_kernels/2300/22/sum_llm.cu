#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(const float *input, float *output, int numElements) {
    // Calculate global index for this thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for partial sums
    __shared__ float partialSum[256]; // Assuming blockDim.x <= 256

    // Initialize partial sum for this thread
    float val = 0.f;

    // Accumulate elements of the array in chunks
    for (int i = index; i < numElements; i += blockDim.x * gridDim.x) {
        val += input[i];
    }

    // Store partial sum in shared memory
    partialSum[threadIdx.x] = val;
    __syncthreads();

    // Reduce partial sums within the block
    if (threadIdx.x == 0) {
        float blockSum = 0.f;
        for (int i = 0; i < blockDim.x; ++i) {
            blockSum += partialSum[i];
        }
        atomicAdd(output, blockSum); // Accumulate block results atomically
    }
}