#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorMul(const float *A, const float *B, float *C, int numElements)
{
    // Use shared memory to improve memory access efficiency
    extern __shared__ float sdata[];

    int tid = threadIdx.x;
    int i = blockDim.x * blockIdx.x + tid;

    // Load data into shared memory
    if (i < numElements) {
        sdata[tid] = A[i] * B[i];
    }

    __syncthreads();

    // Write result back to global memory
    if (i < numElements) {
        C[i] = sdata[tid];
    }
}