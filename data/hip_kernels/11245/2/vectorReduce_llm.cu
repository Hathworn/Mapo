#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorReduce(const float *global_input_data, float *global_output_data, int numElements)
{
    __shared__ float sdata[1024];
    __shared__ int sindice[1024];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (i < numElements) {
        sdata[tid] = global_input_data[i];
        sindice[tid] = tid;
    } else {
        sdata[tid] = FLT_MAX; // Load maximum float value to handle out-of-bounds indices
        sindice[tid] = -1;    // Invalid index for out-of-bounds elements
    }
    __syncthreads();

    // Perform reduction
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && i + s < numElements) {
            if (sdata[tid] > sdata[tid + s]) {
                sdata[tid] = sdata[tid + s];
                sindice[tid] = sindice[tid + s];
            }
        }
        __syncthreads();
    }

    // Output result
    if (tid == 0) {
        global_output_data[blockIdx.x * 2] = sdata[0]; // Use block index for output
        global_output_data[blockIdx.x * 2 + 1] = sindice[0];
    }
}