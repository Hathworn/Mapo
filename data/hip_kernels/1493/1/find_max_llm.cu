#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_max(int* input, int* result, int n)
{
    extern __shared__ int sdata[];  // Use dynamic shared memory
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tx = threadIdx.x;
    int x = (i < n) ? input[i] : -INT_MAX;  // Inline conditional operation

    sdata[tx] = x;
    __syncthreads();
    
    for (unsigned int s = blockDim.x >> 1; s > 0; s >>= 1)
    {
        if (tx < s)
        {
            sdata[tx] = max(sdata[tx], sdata[tx + s]);  // Use max() function
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        result[blockIdx.x] = sdata[0];
    }
}