#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

#define size 10
#define block 10

__global__ void find_max(int* input, int* result, int n)
{
    __shared__ int sdata[block];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int tx = threadIdx.x;
    
    // Initialize shared memory only when in valid range
    if (i<n)
    {
        sdata[tx] = input[i];
    }
    else
    {
        sdata[tx] = -INT_MAX;
    }
    __syncthreads();
    
    // Reduce using loop unrolling
    for (unsigned int s = blockDim.x >> 1; s > 0; s >>= 1)
    {
        if (tx < s)
        {
            int temp = sdata[tx + s];
            sdata[tx] = max(sdata[tx], temp); // Use max function for clarity
        }
        __syncthreads();
    }

    // Write the result of block to global memory
    if (tx == 0)
    {
        result[blockIdx.x] = sdata[0];
    }
}