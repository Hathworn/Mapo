#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nmfcpy(double *mat, double *matcp, int m, int n) 
{
    // Calculate the global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a single check to reduce branching
    int index = row * n + col;
    if (row < m && col < n) 
    {
        // Direct indexing to optimize memory access
        mat[index] = matcp[index];
    }
}