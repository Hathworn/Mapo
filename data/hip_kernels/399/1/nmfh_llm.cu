#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

#define BLOCKSIZE 32

//test code
__global__ void nmfh(float *a, int r, int c, int k, float *w, float *h, float *hcp) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute H
    if (row < k && col < c) {
        float temp = 0.0f;
        float sum1 = 0.0f;
        float sum2 = 0.0f;

        // Optimize inner summation loop for W'A
        #pragma unroll
        for (int i = 0; i < r; i++) {
            sum1 += w[i * k + row] * a[i * c + col];
        }
        temp = h[row * c + col] * sum1;

        // Optimize summation loop for W'WH
        for (int i = 0; i < k; i++) {
            #pragma unroll
            for (int j = 0; j < r; j++) {
                sum2 += w[j * k + row] * w[j * k + i] * h[i * c + col];
            }
        }

        // Remove unnecessary __syncthreads()
        hcp[row * c + col] = temp / sum2;
    }
}