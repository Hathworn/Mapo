#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

#define BLOCKSIZE 32

//test code
__global__ void nmfw(float *a, int r, int c, int k, float *w, float *h, float *wcp)
{
    // Calculate global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds and perform the operation
    if (col < k && row < r) {
        float sum_a = 0.0f;
        float temp_w = w[row * k + col];
        
        // Compute A * H'
        for (int i = 0; i < c; i++) {
            sum_a += a[row * c + i] * h[col * c + i];
        }
        temp_w *= sum_a;

        float sum_whh = 0.0f;
        // Compute WHH'
        for (int i = 0; i < c; i++) {
            for (int j = 0; j < k; j++) {
                sum_whh += w[row * k + j] * h[j * c + i] * h[col * c + i];
            }
        }
        
        // Update WCP value
        wcp[row * k + col] = temp_w / sum_whh;
    }
}
```
