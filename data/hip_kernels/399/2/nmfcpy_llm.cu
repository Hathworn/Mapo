#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

#define BLOCKSIZE 32

// Optimized kernel
__global__ void nmfcpy(float *mat, float *matcp, int m, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Compute global row index
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Compute global column index
    int idx = row * n + col; // Compute linear index

    if (row < m && col < n) {
        mat[idx] = matcp[idx]; // Perform the copy operation
    }
}