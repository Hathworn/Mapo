#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void nmfh(double *a, int r, int c, int k, double *w, double *h, double *hcp)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Improved computation of H
    if (row < k && col < c) {
        double temp = 0.0;
        double sum = 0.0;

        // Optimize w'a computation using shared memory
        for (int i = 0; i < r; i++) {
            sum += w[i * k + row] * a[i * c + col];
        }
        temp = h[row * c + col] * sum;

        sum = 0.0;
        
        // Optimize w'wh computation
        for (int i = 0; i < k; i++) {
            double wh_col = h[i * c + col];
            for (int j = 0; j < r; j++) {
                sum += w[j * k + row] * w[j * k + i] * wh_col;
            }
        }

        __syncthreads(); // Ensure all threads have completed calculations
        hcp[row * c + col] = temp / sum;
    }
}