#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void quad(float *a, int n, float *u, float *v)
{
    int col  = blockIdx.x * blockDim.x + threadIdx.x;
    int row  = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds and compute when col >= row
    if (row < n && col < n && col >= row) {
        float sum = u[col]*a[row*n+col]*u[row];
        
        // Use shared memory to reduce atomic operations
        __shared__ float sharedSum;
        sharedSum = 0.0f;
        __syncthreads();

        // Atomic add to shared memory
        if (col == row) {
            atomicAdd(&sharedSum, sum);
        } else {
            atomicAdd(&sharedSum, 2*sum);
        }
        
        __syncthreads();

        // Atomic add from shared memory to global memory
        if (threadIdx.x == 0 && threadIdx.y == 0) {
            atomicAdd(v, sharedSum);
        }
    }
}