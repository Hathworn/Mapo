#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nmfw(double *a, int r, int c, int k, double *w, double *h, double *wcp)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Compute W
    if (col < k && row < r) {
        double sum_ah = 0.0;
        double sum_whh = 0.0;

        // Calculate ah'
        for (int i = 0; i < c; i++) {
            sum_ah += a[row * c + i] * h[col * c + i];
        }

        double temp = w[row * k + col] * sum_ah;

        // Calculate whh'
        for (int i = 0; i < c; i++) {
            for (int j = 0; j < k; j++) {
                sum_whh += w[row * k + j] * h[j * c + i] * h[col * c + i];
            }
        }

        // Update result
        if (sum_whh != 0.0) {
            wcp[row * k + col] = temp / sum_whh;
        } else {
            wcp[row * k + col] = 0.0; // Handle division by zero
        }
    }
}