#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_RNG_setup(hiprandState *state, unsigned long seed, int N)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure id remains within bounds
    if (id < N) {
        hiprand_init((seed << 20) + id, 0, 0, &state[id]);
    }
}