#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_totalTemp_kernel(int N, double *partialT, double *totalT)
{
    extern __shared__ double T_cache[];
    int tid = threadIdx.x;
    
    // Load data into shared memory efficiently
    T_cache[tid] = (tid < N) ? partialT[tid] : 0.0;
    __syncthreads();
    
    int nTotalThreads = blockDim.x;
    
    // Optimize reduction with loop unrolling; process multiple elements per thread
    for (int stride = nTotalThreads / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            T_cache[tid] += T_cache[tid + stride];
        }
        __syncthreads();
    }
    
    // Only the first thread in block computes and writes the total temperature
    if (tid == 0) {
        double T = T_cache[0];
        T /= (kb * dim * N);  // Compute instantaneous temperature using the Equipartition Theorem
        *totalT = T;
    }
}