#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_reduce_kernel (int N, float * vector, float * sum)
{
    extern __shared__ float partialSum[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int lane = threadIdx.x;

    partialSum[lane] = 0.f;

    __syncthreads();

    for (int i = tid; i < N; i += blockDim.x * gridDim.x)
    {
        partialSum[lane] += vector[i];
    }

    __syncthreads();

    // Reduction using warp shuffles for better performance
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (lane < stride)
        {
            partialSum[lane] += partialSum[lane + stride];
        }
        __syncthreads();  // Synchronize threads within block
    }

    if (lane == 0)
    {
        atomicAdd(sum, partialSum[0]);  // Use atomic operation to safely update global sum
    }
}