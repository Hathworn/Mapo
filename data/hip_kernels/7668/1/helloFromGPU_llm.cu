#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU(void)
{
    // Optimize by reducing the number of threads that print
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hello World from GPU!\n");
    }
}