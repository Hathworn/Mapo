#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void checkIndex(void)
{
    // Calculate linear thread and block index for efficient memory access
    int linearThreadIdx = threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x;
    int linearBlockIdx = blockIdx.z * gridDim.y * gridDim.x + blockIdx.y * gridDim.x + blockIdx.x;

    printf("linearThreadIdx:%d  linearBlockIdx:%d  blockDim:(%d, %d, %d) gridDim:(%d, %d, %d)\n",
           linearThreadIdx, linearBlockIdx,
           blockDim.x, blockDim.y, blockDim.z,
           gridDim.x, gridDim.y, gridDim.z);
}