#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early exit if index exceeds boundary
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Use shared memory for faster access
    __shared__ unsigned int shared_data[BLOCKSIZE / sizeof(unsigned int)];
    
    // Prefetch to shared memory
    for (unsigned int i = threadIdx.x; i < half_count; i += blockDim.x) {
        shared_data[i] = ptr[i];
    }
    __syncthreads();

    // First movement using shared memory
    for (unsigned int i = threadIdx.x; i < half_count - 8; i += blockDim.x) {
        shared_data[i + 8] = shared_data[i + half_count];
    }
    __syncthreads();

    // Second movement using shared memory
    for (unsigned int i = threadIdx.x; i < 8; i += blockDim.x) {
        shared_data[i] = shared_data[half_count - 8 + i];
    }
    __syncthreads();

    // Write back to global memory
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        ptr[i] = shared_data[i];
    }
}