#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    unsigned int i;
    // Calculate the starting index for this block
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* start_ptr = (unsigned int*)_start_ptr;

    // Exit if the pointer exceeds end pointer
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Use unrolled loop for better performance
    #pragma unroll
    for (i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = start_ptr[i];
    }

    return;
}