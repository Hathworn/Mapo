#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    // Calculate the start pointer for this block
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early exit if starting beyond end
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Unroll loop for performance
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = pattern;
    }
}