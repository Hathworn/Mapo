#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Ensure memory write is valid and within bounds.
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Use shared memory or register for intermediate operations if beneficial.
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = pattern;
    }
}