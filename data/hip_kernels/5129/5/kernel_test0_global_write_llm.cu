#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    // Using shared memory to reduce global memory access
    __shared__ unsigned int local_pattern[1];
    local_pattern[0] = 1; // Initial pattern
    
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int* orig_ptr = ptr;

    unsigned long mask = 4;
    *ptr = local_pattern[0];

    while(ptr < end_ptr){
        ptr = (unsigned int*)(((unsigned long)orig_ptr) | mask);
        if (ptr == orig_ptr){
            mask = mask << 1;
            continue;
        }
        if (ptr >= end_ptr){
            break;
        }

        *ptr = local_pattern[0];

        // Shift pattern in shared memory
        local_pattern[0] = local_pattern[0] << 1;
        mask = mask << 1;
    }
    return;
}