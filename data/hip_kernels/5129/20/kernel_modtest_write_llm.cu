#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2) {
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early exit if pointer exceeds end
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Unroll the loop with fixed offsets for improved performance
    #pragma unroll
    for (unsigned int i = offset; i < BLOCKSIZE / sizeof(unsigned int); i += MOD_SZ) {
        ptr[i] = p1;
    }

    // Combine the two loops to reduce overhead
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        if (i % MOD_SZ != offset) {
            ptr[i] = p2;
        }
    }
}