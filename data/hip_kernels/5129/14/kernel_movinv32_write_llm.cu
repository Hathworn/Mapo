#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Exit early if the starting address is beyond the end pointer
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int k = offset;
    unsigned pat = pattern;
    
    // Utilize threadIdx.x to avoid loop when accessing array elements
    unsigned int i = threadIdx.x * (BLOCKSIZE / sizeof(unsigned int) / blockDim.x);
    if (i < BLOCKSIZE / sizeof(unsigned int)) {
        ptr[i] = pat;
        k += i;
        
        // Calculate new pattern
        if (k >= 32) {
            pat = lb;
        } else {
            pat = (pat << i) | ((1 << i) - 1) & sval;
        }
    }
}