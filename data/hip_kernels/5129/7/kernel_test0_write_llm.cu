#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    // Use blockIdx and threadIdx to calculate the starting memory position
    unsigned int* orig_ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE + threadIdx.x * sizeof(unsigned int));
    unsigned int* ptr = orig_ptr;

    // Guard against out-of-bounds access
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int* block_end = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE + BLOCKSIZE);

    // Initialize pattern and mask
    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Write initial pattern
    *ptr = pattern;

    // Iterate with mask to populate the block
    while (ptr < block_end) {
        ptr = (unsigned int*) (((unsigned long)orig_ptr) | mask);
        
        // Ensure ptr is not reverting to orig_ptr which causes infinite loop
        if (ptr == orig_ptr) {
            mask <<= 1;
            continue;
        }

        // Ensure we don't write beyond the block's end
        if (ptr >= block_end) {
            break;
        }

        // Update memory with current pattern
        *ptr = pattern;

        // Update pattern and mask for next iteration
        pattern <<= 1;
        mask <<= 1;
    }
}