#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test5_init(char* _ptr, char* end_ptr)
{
    // Calculate global thread index
    unsigned int global_tid = blockIdx.x * BLOCKSIZE + threadIdx.x;

    // Convert starting pointer to unsigned int
    unsigned int* ptr = (unsigned int*)_ptr;

    // Calculate the number of unsigned ints per block
    unsigned int num_uints_per_block = BLOCKSIZE / sizeof(unsigned int);

    // Compute starting position based on global thread index
    unsigned int start_idx = global_tid * num_uints_per_block;

    // Ensure the starting position is within bounds
    if ((ptr + start_idx) >= (unsigned int*)end_ptr) {
        return;
    }

    // Initialize p1
    unsigned int p1 = 1;

    // Unrolled loop for initialization in sets of 16
    for (unsigned int i = start_idx; i < start_idx + num_uints_per_block; i += 16) {
        unsigned int p2 = ~p1;

        // Initialize pattern
        ptr[i] = p1;
        ptr[i + 1] = p1;
        ptr[i + 2] = p2;
        ptr[i + 3] = p2;
        ptr[i + 4] = p1;
        ptr[i + 5] = p1;
        ptr[i + 6] = p2;
        ptr[i + 7] = p2;
        ptr[i + 8] = p1;
        ptr[i + 9] = p1;
        ptr[i + 10] = p2;
        ptr[i + 11] = p2;
        ptr[i + 12] = p1;
        ptr[i + 13] = p1;
        ptr[i + 14] = p2;
        ptr[i + 15] = p2;

        // Update p1 and reset if overflow
        p1 <<= 1;
        if (p1 == 0) {
            p1 = 1;
        }
    }
}