#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    // Calculate starting pointer for this block
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check if pointer is within the valid range
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }

    // Use threadIdx.x to write data in parallel
    int idx = threadIdx.x;
    int stride = blockDim.x;
    
    // Loop through data in steps of warp size for coalesced memory access
    for (int i = idx; i < BLOCKSIZE / sizeof(unsigned long); i += stride) {
        ptr[i] = (unsigned long)&ptr[i];
    }
}