#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void d_MM(float *a, float *b, float *c, int wA, int wB, int hA)
{
    // global index
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;  // col
    int gidy = blockDim.y * blockIdx.y + threadIdx.y;  // row

    // Check valid global indices
    if (gidx < wB && gidy < hA)
    {
        float sum = 0.f;
        int aIndex = gidy * wA;
        int bIndex = gidx;
        // Optimize for-loop to minimize index calculation
        for (int k = 0; k < wA; ++k)
        {
            sum += a[aIndex + k] * b[k * wB + bIndex];
        }
        c[gidy * wB + gidx] = sum;
    }
}