#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuMultOpti( int *a, int *b, int *c, int wA, int wB, int hA)
{
#define blockTile 16
/* Blocksize is 16x16 */
/* Allocate shared memory */
__shared__ int aBlock[blockTile][blockTile];
__shared__ int bBlock[blockTile][blockTile];

/* Calculate global index X, Y */
int gidx = blockDim.x * blockIdx.x + threadIdx.x;  // column
int gidy = blockDim.y * blockIdx.y + threadIdx.y;  // row

/* Initialize sum */
int sum = 0;
/* Loop over the tiles */
for (int m = 0; m < (wA - 1) / blockTile + 1; ++m) {
    /* Collaborative loading of a and b into shared memory */
    if (m * blockTile + threadIdx.x < wA && gidy < hA) {
        aBlock[threadIdx.y][threadIdx.x] = a[gidy * wA + m * blockTile + threadIdx.x];
    } else {
        aBlock[threadIdx.y][threadIdx.x] = 0;
    }

    if (m * blockTile + threadIdx.y < wA && gidx < wB) {
        bBlock[threadIdx.y][threadIdx.x] = b[(m * blockTile + threadIdx.y) * wB + gidx];
    } else {
        bBlock[threadIdx.y][threadIdx.x] = 0;
    }

    /* Make sure all threads have cached the memory */
    __syncthreads();

    for (int k = 0; k < blockTile; ++k) {
        sum += aBlock[threadIdx.y][k] * bBlock[k][threadIdx.x];
    }

    /* Synchronize to make sure all threads are done computing */
    __syncthreads();
}

/* Write result back to global memory */
if (gidx < wB && gidy < hA) {
    c[gidy * wB + gidx] = sum;
}
}