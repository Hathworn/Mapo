#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define blockTile 16

__global__ void d_MM_OPT(float *a, float *b, float *c, int wA, int wB, int hA)
{
    __shared__ float aBlock[blockTile][blockTile];
    __shared__ float bBlock[blockTile][blockTile];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int gx = blockDim.x * bx + tx;  // column
    int gy = blockDim.y * by + ty;  // row

    int a0 = wA * blockTile * by;
    int aZ = a0 + wA - 1;
    int b0 = blockTile * bx;
    int bD = blockTile * wB;
    float sum = 0.f;

    // Loop over tiles and accumulate results asynchronously
    for(int aI = a0, bI = b0; aI <= aZ; aI += blockTile, bI += bD)
    {
        // Optimize memory access using shared memory
        if ((aI + ty * wA + tx) < (wA * hA)) // Check if within bounds
            aBlock[ty][tx] = a[aI + ty * wA + tx];
        else 
            aBlock[ty][tx] = 0;  // Avoid out-of-bound access

        if ((bI + ty * wB + tx) < (hA * wB)) // Check if within bounds
            bBlock[ty][tx] = b[bI + ty * wB + tx];
        else 
            bBlock[ty][tx] = 0;  // Avoid out-of-bound access

        __syncthreads();

        for(int k = 0; k < blockTile; k++)
        {
            sum += aBlock[ty][k] * bBlock[k][tx];
        }
        __syncthreads(); // Ensure all threads have completed the current tile
    }

    if (gx < wB && gy < hA) // Ensure result is within bounds
        c[gy * wB + gx] = sum;
}