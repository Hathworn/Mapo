#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuMult(int *a, int *b, int *c, int wA, int wB, int hA)
{
    // Shared memory for tile of Matrix A and B
    extern __shared__ int sharedMem[];
    int* tileA = sharedMem;
    int* tileB = sharedMem + blockDim.y * wA;

    // Block index
    int blockRow = blockIdx.y * blockDim.y;
    int blockCol = blockIdx.x * blockDim.x;

    // Thread index
    int row = blockRow + threadIdx.y;
    int col = blockCol + threadIdx.x;

    int sum = 0;

    // Loop over tiles of the input in steps of blockDim.x and blockDim.y
    for (int m = 0; m < (wA + blockDim.x - 1) / blockDim.x; ++m) {
        // Load tiles into shared memory
        if (row < hA && m * blockDim.x + threadIdx.x < wA)
            tileA[threadIdx.y * wA + threadIdx.x] = a[row * wA + m * blockDim.x + threadIdx.x];
        else
            tileA[threadIdx.y * wA + threadIdx.x] = 0;
        
        if (col < wB && m * blockDim.y + threadIdx.y < wA)
            tileB[threadIdx.y * wB + threadIdx.x] = b[(m * blockDim.y + threadIdx.y) * wB + col];
        else
            tileB[threadIdx.y * wB + threadIdx.x] = 0;

        __syncthreads();

        // Perform multiplication of the partial tiles
        for (int k = 0; k < blockDim.x; ++k) {
            sum += tileA[threadIdx.y * wA + k] * tileB[k * wB + threadIdx.x];
        }

        __syncthreads();
    }

    // Write back the result
    if (row < hA && col < wB) {
        c[row * wB + col] = sum;
    }
}