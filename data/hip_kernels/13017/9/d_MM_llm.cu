#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void d_MM(float *a, float *b, float *c, int wA, int wB, int hA)
{
    // global index
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;  // col
    int gidy = blockDim.y * blockIdx.y + threadIdx.y;  // row

    if (gidx < wB && gidy < hA)
    {
        float sum = 0.f;
        // unroll loop for better performance
        for (int k = 0; k < wA; k += 4)
        {
            sum += a[gidy * wA + k] * b[k * wB + gidx];
            sum += a[gidy * wA + k + 1] * b[(k + 1) * wB + gidx];
            sum += a[gidy * wA + k + 2] * b[(k + 2) * wB + gidx];
            sum += a[gidy * wA + k + 3] * b[(k + 3) * wB + gidx];
        }
        c[gidy * wB + gidx] = sum;
    }
}