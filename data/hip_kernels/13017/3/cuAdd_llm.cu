#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuAdd(int *a, int *b, int *c, int N)
{
    // Calculate global offset
    int offset = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x; 
    if (offset < N)
    {
        c[offset] = a[offset] + b[offset]; // Perform element-wise addition
    }
}