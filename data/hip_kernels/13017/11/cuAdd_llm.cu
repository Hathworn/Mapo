#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuAdd(int *a, int *b, int *c, int N)
{
    // Get 1D global index
    int offset = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure thread is within bounds
    if (offset < N)
    {
        // Directly perform addition
        c[offset] = a[offset] + b[offset];
    }
}