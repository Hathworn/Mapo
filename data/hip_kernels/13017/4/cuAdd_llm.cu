#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized version of the cuAdd kernel function
__global__ void cuAdd(int *a, int *b, int *c, int N)
{
    // Calculate global index using grid-stride loop for improved occupancy
    for(int offset = blockDim.x * blockIdx.x + threadIdx.x; offset < N; offset += blockDim.x * gridDim.x)
    {
        c[offset] = a[offset] + b[offset];
    }
}