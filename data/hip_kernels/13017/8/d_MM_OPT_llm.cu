#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void d_MM_OPT(float *a, float *b, float *c, int wA, int wB, int hA) 
{
#define blockTile 16

    __shared__ float aBlock[blockTile][blockTile];
    __shared__ float bBlock[blockTile][blockTile];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int gx = blockDim.x * bx + tx;  // column
    int gy = blockDim.y * by + ty;  // row

    int a0 = wA * blockTile * by;
    int aD = blockTile;
    int b0 = blockTile * bx;
    int bD = blockTile * wB;

    float sum = 0.f;

    for (int aI = a0, bI = b0; aI <= a0 + wA - 1; aI += aD, bI += bD) 
    {
        // Load matrices into shared memory and prevent out-of-bounds access
        if (gy < hA && gx < wA) 
        {
            aBlock[ty][tx] = a[aI + ty * wA + tx];
        }
        else 
        {
            aBlock[ty][tx] = 0.0f;
        }

        if (gy < wB && gx < hA) 
        {
            bBlock[ty][tx] = b[bI + ty * wB + tx];
        }
        else 
        {
            bBlock[ty][tx] = 0.0f;
        }

        __syncthreads();

        // Compute partial product
        for (int k = 0; k < blockTile; ++k) 
        {
            sum += aBlock[ty][k] * bBlock[k][tx];
        }
        __syncthreads();
    }

    // Write result to global memory only if indices are valid
    if (gy < hA && gx < wB) 
    {
        c[gy * wB + gx] = sum;
    }
}