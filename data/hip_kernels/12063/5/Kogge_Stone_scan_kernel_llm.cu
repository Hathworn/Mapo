#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Kogge_Stone_scan_kernel(float *X, float *Y, int InputSize)
{
    __shared__ float XY[SECTION_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < InputSize) {
        XY[threadIdx.x] = X[i];
    } else {
        XY[threadIdx.x] = 0.0f; // Initialize out-of-bounds elements
    }

    // Perform iterative scan on XY
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        float in = 0.0f; // Pre-initialize 'in' variable
        __syncthreads();
        if (threadIdx.x >= stride) {
            in = XY[threadIdx.x - stride];
        }
        __syncthreads();
        XY[threadIdx.x] += in; // Move addition outside of condition
    }

    __syncthreads();
    if (i < InputSize) { // Conditional write to global memory
        Y[i] = XY[threadIdx.x];
    }
}