#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void efficient_Kogge_Stone_scan_kernel(float *X, float *Y, int InputSize) {
    __shared__ float XY[SECTION_SIZE];
    __shared__ float AUS[BLOCK_DIM];

    // Coalesced load of input into shared memory
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;
    int stride = blockDim.x;

    for (int j = 0; j < SECTION_SIZE; j += stride) {
        int shared_idx = tid + j;
        if (shared_idx < InputSize) {
            XY[shared_idx] = X[idx + j];
        }
    }
    __syncthreads();

    // PHASE 1: Inclusive scan within each thread's subsection
    for (int j = 1; j < SUBSECTION_SIZE; j++) {
        int subsection_idx = tid * SUBSECTION_SIZE + j;
        XY[subsection_idx] += XY[subsection_idx - 1];
    }
    __syncthreads();

    // PHASE 2: Iteratively Kogge-Stone scan across subsections
    AUS[tid] = XY[tid * SUBSECTION_SIZE + SUBSECTION_SIZE - 1];
    float in;
    for (unsigned int s = 1; s < stride; s *= 2) {
        __syncthreads();
        if (tid >= s) {
            in = AUS[tid - s];
        }
        __syncthreads();
        if (tid >= s) {
            AUS[tid] += in;
        }
    }
    __syncthreads();

    // PHASE 3: Add scanned results to each subsection
    if (tid > 0) {
        float offset = AUS[tid - 1];
        for (unsigned int j = 0; j < SUBSECTION_SIZE; j++) {
            XY[tid * SUBSECTION_SIZE + j] += offset;
        }
    }
    __syncthreads();

    // Store results into output
    for (int j = 0; j < SECTION_SIZE; j += stride) {
        int shared_idx = tid + j;
        if (shared_idx < InputSize) {
            Y[idx + j] = XY[shared_idx];
        }
    }
}