#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void partialSumKernel(int *X, int N)
{
    __shared__ int partialSum[BLOCK_SIZE * 2]; // Allocate for full range
    int tx = threadIdx.x;
    int i = blockIdx.x * blockDim.x * 2 + tx; // Process two elements per thread

    // Load data into shared memory with boundary checking
    partialSum[tx] = (i < N) ? X[i] : 0;
    partialSum[tx + blockDim.x] = (i + blockDim.x < N) ? X[i + blockDim.x] : 0;

    for (int stride = blockDim.x; stride > 0; stride >>= 1) {
        __syncthreads();
        if (tx < stride) {
            partialSum[tx] += partialSum[tx + stride];
        }
    }

    if (tx == 0) {
        X[blockIdx.x] = partialSum[0]; 
    }
}