#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Brent_Kung_scan_kernel(float *X, float *Y, int InputSize)
{
    __shared__ float XY[SECTION_SIZE];
    int i = 2 * blockIdx.x * blockDim.x + threadIdx.x;

    // Load input to shared memory with bounds checks
    if (i < InputSize) XY[threadIdx.x] = X[i];
    else XY[threadIdx.x] = 0;
    if (i + blockDim.x < InputSize) XY[threadIdx.x + blockDim.x] = X[i + blockDim.x];
    else XY[threadIdx.x + blockDim.x] = 0;

    // Up-sweep (reduction) phase
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * 2 * stride - 1;
        if (index < SECTION_SIZE) {
            XY[index] += XY[index - stride];
        }
    }

    // Down-sweep phase
    for (int stride = SECTION_SIZE / 4; stride > 0; stride /= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < SECTION_SIZE) {
            XY[index + stride] += XY[index];
        }
    }

    __syncthreads();

    // Write the results back to global memory with bounds checks
    if (i < InputSize) Y[i] = XY[threadIdx.x];
    if (i + blockDim.x < InputSize) Y[i + blockDim.x] = XY[threadIdx.x + blockDim.x];
}