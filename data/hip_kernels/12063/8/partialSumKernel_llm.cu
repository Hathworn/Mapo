#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void partialSumKernel(int *X, int N)
{
    __shared__ int partialSum[2 * BLOCK_SIZE];
    int tx = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tx;

    // Optimize load by adding a conditional check
    partialSum[tx] = (i < N) ? X[i] : 0;
    partialSum[tx + blockDim.x] = 0;

    __syncthreads(); // Ensure all loads are completed
    
    // Use loop unrolling to reduce the number of iterations
    for (int stride = blockDim.x; stride > 0; stride >>= 1) {
        if (tx < stride) {
            partialSum[tx] += partialSum[tx + stride];
        }
        __syncthreads(); // Ensure all adds are completed before next iteration
    }

    // Write back the result to global memory
    if (tx == 0) {
        X[blockIdx.x] = partialSum[0];
    }
}