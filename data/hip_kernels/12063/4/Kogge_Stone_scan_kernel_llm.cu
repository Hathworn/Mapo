#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Kogge_Stone_scan_kernel(float *X, float *Y, int InputSize)
{
    __shared__ float XY[SECTION_SIZE];
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize shared memory and handle boundaries more efficiently
    XY[threadIdx.x] = (i > 0 && i <= InputSize) ? X[i - 1] : 0;
    
    // Perform iterative exclusive scan on XY
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        __syncthreads();  // Moved to the start of the loop for proper data synchronization
        float t = (threadIdx.x >= stride) ? XY[threadIdx.x - stride] : 0;
        __syncthreads();  // Ensure all reads are completed
        XY[threadIdx.x] += t;
    }
    
    // Write result to global memory if within bounds
    if (i < InputSize)
    {
        Y[i] = XY[threadIdx.x];
    }
}