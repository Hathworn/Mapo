#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hierarchical_scan_kernel_phase1(int *X, int *Y, int *S) {
    __shared__ int XY[SECTION_SIZE];
    __shared__ int AUS[BLOCK_DIM];
    int tx = threadIdx.x, bx = blockIdx.x;
    int i = bx * SECTION_SIZE + tx;

    if (i < INPUT_SIZE) {
        // Load data collaboratively, ensuring coalesced access
        for (int j = 0; j < SECTION_SIZE; j += BLOCK_DIM) {
            XY[tx + j] = X[i + j];
        }
        __syncthreads();

        // Phase 1: Perform an inclusive scan on each subsection
        for (int j = 1; j < SUBSECTION_SIZE; j++) {
            XY[tx * SUBSECTION_SIZE + j] += XY[tx * SUBSECTION_SIZE + j - 1];
        }
        __syncthreads();

        // Phase 2: Iteratively perform a Kogge-Stone scan on the last elements of subsections
        AUS[tx] = XY[(tx + 1) * SUBSECTION_SIZE - 1];
        for (unsigned int stride = 1; stride < BLOCK_DIM; stride *= 2) {
            int in = 0; // Use a local variable for temporary storage
            __syncthreads();
            if (tx >= stride) {
                in = AUS[tx - stride];
            }
            __syncthreads();
            if (tx >= stride) {
                AUS[tx] += in;
            }
        }
        __syncthreads();

        // Phase 3: Add the prefix sum of the previous subsections
        if (tx > 0) {
            int addVal = AUS[tx - 1]; // Store the value to add in a local variable
            for (unsigned int stride = 0; stride < SUBSECTION_SIZE; stride++) {
                XY[tx * SUBSECTION_SIZE + stride] += addVal;
            }
        }
        __syncthreads();

        // Store the result into the output vector
        for (int j = 0; j < SECTION_SIZE; j += BLOCK_DIM) {
            Y[i + j] = XY[tx + j];
        }

        // The last thread writes the output value of the last element to the block index position of S
        if (tx == BLOCK_DIM - 1) {
            S[bx] = XY[SECTION_SIZE - 1];
        }
    }
}