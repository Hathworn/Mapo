#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hierarchical_scan_kernel_phase3(int *S, int *Y) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int i = bx * SECTION_SIZE + tx;

    // Utilize shared memory for improved memory access pattern
    __shared__ int shared_S;
    if (tx == 0) {
        shared_S = bx > 0 ? S[bx - 1] : 0;
    }
    __syncthreads();

    // Loop unrolling for performance optimization
    for (int j = 0; j < SECTION_SIZE; j += BLOCK_DIM) {
        Y[i + j] += shared_S;
    }
}