#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_cmp_kernel(std::size_t n, int* aptr, int* bptr, int* rptr) {
    // Calculate global index
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use shared memory to reduce global memory access
    __shared__ int flag;
    if (threadIdx.x == 0) flag = 0;
    __syncthreads();
    
    // Perform comparison and set flag if condition is met
    if (i < n && aptr[i] < bptr[i]) atomicExch(&flag, 1);
    __syncthreads();
    
    // Store result if any thread found aptr[i] < bptr[i]
    if (threadIdx.x == 0 && flag == 1) *rptr = 1;
}