#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int test()
{
    return 10;
}

__global__ void testDrive()
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Load device function result into shared memory for reuse
    __shared__ int num;
    if (threadIdx.x == 0)
    {
        num = test();
    }
    __syncthreads();

    printf("%d\n", index);
    printf("num = %d\n", num);
}