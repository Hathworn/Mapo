#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Add(float *A, int size)
{
    // Cache grid and block dimensions
    const unsigned int block_size = blockDim.x;
    const unsigned int grid_size = gridDim.x;
    const unsigned int total_threads = block_size * grid_size;

    // Calculate global thread index
    const int global_idx = blockIdx.x * block_size + threadIdx.x;

    // Loop over elements with stride equal to total number of threads
    for (unsigned int i = global_idx; i < size; i += total_threads) {
        A[i] += A[i];
    }
}