#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stdp_kernel( float *weight, int weight_size_0, int weight_size_1, int weight_size_2, int weight_size_3, float *output_spike, int output_spike_size_0, int output_spike_size_1, int output_spike_size_2, int output_spike_size_3, float *history, float *weight_update)
{
    // Constants for index calculations
    const int offset = weight_size_2 / 2;
    const int batch_id = blockIdx.x;
    const int channel_id = threadIdx.x;
    const int height_id = blockIdx.y;
    const int width_id = blockIdx.z;
    const int linear_idx = (batch_id * (output_spike_size_1 * output_spike_size_2 * output_spike_size_3)) \
                          + (channel_id * (output_spike_size_2 * output_spike_size_3)) \
                          + (height_id * (output_spike_size_3)) \
                          + width_id;

    // Early exit if output spike is not 1
    if (output_spike[linear_idx] != 1.0f) return;

    const int filter_index = channel_id * weight_size_1 * weight_size_2 * weight_size_3;
    const int delta_index_const = batch_id * weight_size_0 * weight_size_1 * weight_size_2 * weight_size_3;

    // Loop through weight dimensions
    for (int l = 0; l < weight_size_1; l++)
    {
        int history_pivot = (batch_id * (weight_size_1 * output_spike_size_2 * output_spike_size_3)) \
                          + (l * (output_spike_size_2 * output_spike_size_3));
        int filter_2d_index = filter_index + (l * weight_size_2 * weight_size_3);

        for (int i = 0; i < weight_size_2; i++)
        {
            int filter_1d_index = filter_2d_index + (i * weight_size_3);
            int history_index1 = height_id + i - offset;

            if (history_index1 < 0 || history_index1 >= output_spike_size_2) continue;

            int history_1d_index = history_pivot + (history_index1 * output_spike_size_3);

            for (int j = 0; j < weight_size_3; j++)
            {
                int w_index = filter_1d_index + j;
                int deltaW_index = delta_index_const + w_index;
                int history_index2 = width_id + j - offset;

                if (history_index2 < 0 || history_index2 >= output_spike_size_3) continue;

                int history_index = history_1d_index + history_index2;
                float input_tmp = history[history_index];

                // Update weight change based on input history
                weight_update[deltaW_index] = (weight[w_index] * (1 - weight[w_index])) * (input_tmp != 0.0f ? 1 : -1);
            }
        }
    }
}