#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stdp_kernel( float *weight, int weight_size_0, int weight_size_1, int weight_size_2, int weight_size_3, float *output_spike, int output_spike_size_0, int output_spike_size_1, int output_spike_size_2, int output_spike_size_3, float *history, float *weight_update)
{
    // Calculate grid and thread indices
    const int batch_id = blockIdx.x;
    const int channel_id = threadIdx.x;
    const int height_id = blockIdx.y;
    const int width_id = blockIdx.z;

    // Calculate the linear index for output_spike
    const int linear_idx = (batch_id * output_spike_size_1 * output_spike_size_2 * output_spike_size_3) 
                            + (channel_id * output_spike_size_2 * output_spike_size_3) 
                            + (height_id * output_spike_size_3) 
                            + width_id;

    // Check the spike condition
    if (output_spike[linear_idx] != 1.0f) return;

    // Calculate some constants
    const int offset = weight_size_2 / 2;
    const int filter_index = channel_id * weight_size_1 * weight_size_2 * weight_size_3;
    const int delta_index_const = batch_id * weight_size_0 * weight_size_1 * weight_size_2 * weight_size_3;

    // Loop unrolling to maximize performance
    for (int l = 0; l < weight_size_1; l++) {
        int history_pivot = batch_id * weight_size_1 * output_spike_size_2 * output_spike_size_3 
                            + l * output_spike_size_2 * output_spike_size_3;
        int filter_2d_index = filter_index + l * weight_size_2 * weight_size_3;

        for (int i = 0; i < weight_size_2; i++) {
            int filter_1d_index = filter_2d_index + i * weight_size_3;
            int history_index1 = height_id + i - offset;
            
            // Skip invalid history indices
            if (history_index1 < 0 || history_index1 >= output_spike_size_2) continue;
            int history_1d_index = history_pivot + history_index1 * output_spike_size_3;

            for (int j = 0; j < weight_size_3; j++) {
                int history_index2 = width_id + j - offset;
                // Skip invalid history indices
                if (history_index2 < 0 || history_index2 >= output_spike_size_3) continue;
                int history_index = history_1d_index + history_index2;
                int w_index = filter_1d_index + j;
                int deltaW_index = delta_index_const + w_index;

                float input_tmp = history[history_index];
                // Compute weight update
                weight_update[deltaW_index] = (weight[w_index] * (1 - weight[w_index])) 
                                            * ((input_tmp != 0.0f) - (input_tmp == 0.0f));
            }
        }
    }
}