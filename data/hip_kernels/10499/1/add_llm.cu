#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void add(int n, float *a, float *sum)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // Use shared memory to cache partial sums
    extern __shared__ float s_sum[];
    
    if (i < n)
    {
        float temp_sum = 0.0f;

        // Use a single loop to accumulate the sum
        for (int j = 0; j < n; j++)
        {
            temp_sum += a[i * n + j];
        }

        s_sum[threadIdx.x] = temp_sum;
        __syncthreads();

        // Aggregate results from shared memory
        if (threadIdx.x == 0) {
            float block_sum = 0.0f;
            for (int idx = 0; idx < blockDim.x; ++idx) {
                block_sum += s_sum[idx];
            }
            atomicAdd(&sum[blockIdx.x], block_sum);
        }
    }
}