#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void bubble(unsigned int length, unsigned int parity, float* tab)
{
    // Calculate global thread ID
    int index = 2 * (threadIdx.x + blockDim.x * blockIdx.x);
    int leftElementID = index + parity;
    int rightElementID = index + parity + 1;

    // Check bounds and swap if out of order
    if (rightElementID < length)
    {
        float l = tab[leftElementID];
        float r = tab[rightElementID];
        if (r < l)
        {
            // Use temporary variable for swapping to avoid redundant load/store
            tab[leftElementID] = r;
            tab[rightElementID] = l;
        }
    }
}