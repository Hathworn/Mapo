#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void multiply(int n, float *a, float *b, float *sum)
{
    // Calculate thread index for current thread in grid.
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure index is within bounds to avoid invalid memory access.
    if (i < n)
    {
        sum[i] = a[i] * b[i]; // Perform element-wise multiplication.
    }
}