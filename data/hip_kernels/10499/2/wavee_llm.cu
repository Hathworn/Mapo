#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void wavee(int* tab, unsigned int rowSize, unsigned int centerX, unsigned int centerY, float A, float lambda, float time, float fi, unsigned int N)
{
    // Calculate the global index.
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within the bounds.
    if (index < N) {
        // Calculate 2D grid coordinates.
        int w = index / rowSize;
        int h = index % rowSize;

        // Calculate dx and dy in a single expression.
        float dx = static_cast<float>(abs(centerX - w));
        float dy = static_cast<float>(abs(centerY - h));

        // Use built-in functions for distance computation.
        float distance = sqrtf(dx * dx + dy * dy);

        // Precompute constant values.
        const float pi = 3.1415f;
        const float v = 1.0f;
        const float T = lambda / v;
        const float ww = 2.0f * pi / T;
        const float k = 2.0f * pi / lambda;

        // Compute the wave equation.
        float f = A * sinf(ww * time - k * distance + fi);

        // Calculate the result and store it.
        float res = f * 127 + 127;
        tab[index] = static_cast<int>(res);
    }
}