#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_cmp_kernel(std::size_t n, int* aptr, int* bptr, int* rptr) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // Use shared memory for result
    __shared__ int local_result;
    if (threadIdx.x == 0) local_result = 0;

    int cmp = (i < n) ? (aptr[i] < bptr[i]) : 0;

    // Cooperative reduction within block
    atomicOr(&local_result, cmp);

    // Ensure all threads have written to local_result
    __syncthreads();

    // Write to global result by one thread per block
    if (threadIdx.x == 0 && local_result) atomicOr(rptr, 1);
}