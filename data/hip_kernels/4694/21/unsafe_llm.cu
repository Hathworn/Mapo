#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unsafe(int *shared_var, int iters)
{
    // Allocate a register for local copy
    int local_copy = *shared_var;
    
    for (int i = 0; i < iters; i++)
    {
        // Increment local copy
        local_copy += 1;
    }
    
    // Write back to shared variable once
    *shared_var = local_copy;
}