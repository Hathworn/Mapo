#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_diagonal_row(int *mat, int *transpose, int nx, int ny)
{
    // Calculate index using threadIdx and blockIdx
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure index is within matrix bounds before transposing
    if (ix < nx && iy < ny)
    {
        transpose[iy * nx + ix] = mat[ix * ny + iy];
    }
}