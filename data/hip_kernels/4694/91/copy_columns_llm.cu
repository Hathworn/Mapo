#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_columns(float* input, float* output, const int nx, const int ny)
{
    // Calculate a single linear index to improve memory access coalescing
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int i = idx; i < nx * ny; i += stride)
    {
        output[i] = input[i];
    }
}