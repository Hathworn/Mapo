#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction_kernel_interleaved_warp_unrolling8_1(int *input, int *temp_array, int size)
{
    int tid = threadIdx.x;
    int index = blockDim.x * blockIdx.x * 8 + tid;
    int *i_data = input + blockDim.x * blockIdx.x * 8;

    if ((index + 7 * blockDim.x) < size)
    {
        // Use local variables for reduction to facilitate optimized memory access
        int sum = input[index] + input[index + blockDim.x] + input[index + 2 * blockDim.x]
                + input[index + 3 * blockDim.x] + input[index + 4 * blockDim.x]
                + input[index + 5 * blockDim.x] + input[index + 6 * blockDim.x]
                + input[index + 7 * blockDim.x];

        input[index] = sum;
    }

    __syncthreads();

    // Unroll the loop to optimize inter-thread communication
    for (int offset = blockDim.x / 2; offset >= 64; offset /= 2)
    {
        if (tid < offset)
        {
            i_data[tid] += i_data[tid + offset];
        }
        __syncthreads();
    }

    // Use warp-level operations without __syncthreads for efficiency
    if (tid < 32)
    {
        volatile int *vsmem = i_data;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write the result of this block to the output array
    if (tid == 0)
    {
        temp_array[blockIdx.x] = i_data[0];
    }
}