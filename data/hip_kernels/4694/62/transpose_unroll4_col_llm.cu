#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_unroll4_col(int * mat, int * transpose, int nx, int ny)
{
    // Calculate global thread position
    int ix = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Linearized indices for read and write
    int ti = iy * nx + ix;
    int to = ix * ny + iy;

    // Check bounds only once to ensure all required elements are within range
    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        // Coalesced access for optimized memory read
        transpose[ti] = mat[to];
        transpose[ti + blockDim.x] = mat[to + blockDim.x * ny];
        transpose[ti + 2 * blockDim.x] = mat[to + 2 * blockDim.x * ny];
        transpose[ti + 3 * blockDim.x] = mat[to + 3 * blockDim.x * ny];
    }
}