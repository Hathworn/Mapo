#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_rows(float* input, float* output, const int nx, const int ny)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the linear index for 1D grid traversal
    int index = ix + blockIdx.y * blockDim.y * nx + threadIdx.y * nx;

    if (ix < nx && index < nx * ny) // Single condition check
    {
        output[index] = input[index]; // Use linear index for both input and output
    }
}