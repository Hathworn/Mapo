#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction_neighbored_pairs_improved_1( int * int_array, int * temp_array, int size)
{
    int tid = threadIdx.x;
    int gid = blockDim.x * blockIdx.x + threadIdx.x;

    // local data block pointer
    int * i_data = int_array + blockDim.x * blockIdx.x;

    if (gid >= size) // Fix boundary check
        return;

    // Use shared memory to optimize global memory access
    extern __shared__ int shared_data[];
    shared_data[tid] = (gid < size) ? int_array[gid] : 0;
    __syncthreads();

    for (int offset = 1; offset <= blockDim.x / 2; offset *= 2)
    {
        int index = 2 * offset * tid;
        if (index < blockDim.x)
        {
            // Operate on shared memory
            shared_data[index] += shared_data[index + offset];
        }
        __syncthreads();
    }

    if (tid == 0)
    {
        temp_array[blockIdx.x] = shared_data[0]; // Write result from shared memory
    }
}