#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cb_k2()
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use a single thread for printing to reduce overhead
    if (gid < 1)
    {
        printf("This is a test 2 \n");
    }
}