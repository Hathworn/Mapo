#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyRow_1(float * __restrict__ out, const float * __restrict__ in, const int nx, const int ny)
{
    // Calculate global thread ID
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Use shared memory for tiles (if applicable)
    extern __shared__ float tile[];
    unsigned int tid = threadIdx.y * blockDim.x + threadIdx.x;

    // Improve memory access pattern and coalescing
    if (ix < nx && iy < ny)
    {
        tile[tid] = in[iy * nx + ix];
        __syncthreads(); // Ensure all threads have written their data

        // Write back to global memory
        out[iy * nx + ix] = tile[tid];
    }
}