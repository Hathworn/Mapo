#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_arrays_2Dgrid_2Dblock(float* a, float* b, float* c, int nx, int ny)
{
    // Calculate global thread ID
    int gidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gidy = blockIdx.y * blockDim.y + threadIdx.y;

    // Use grid-stride loop for better performance
    for (int y = gidy; y < ny; y += gridDim.y * blockDim.y) {
        for (int x = gidx; x < nx; x += gridDim.x * blockDim.x) {
            int gid = y * nx + x;
            c[gid] = a[gid] + b[gid];
        }
    }
}