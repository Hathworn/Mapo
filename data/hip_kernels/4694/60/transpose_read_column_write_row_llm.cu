#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_read_column_write_row(int *mat, int *transpose, int nx, int ny)
{
    // Use shared memory for better performance
    __shared__ int tile[32][32 + 1]; // Add padding to avoid bank conflicts

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Transpose using shared memory to reduce global memory accesses
    if (ix < nx && iy < ny)
    {
        tile[threadIdx.y][threadIdx.x] = mat[ix * ny + iy];
    }

    __syncthreads();

    ix = blockIdx.y * blockDim.y + threadIdx.x; // Transpose block offset
    iy = blockIdx.x * blockDim.x + threadIdx.y;

    if (ix < ny && iy < nx)
    {
        transpose[iy * ny + ix] = tile[threadIdx.x][threadIdx.y];
    }
}