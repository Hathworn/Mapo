#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_cuda()
{
    // Use block and thread indices to manage work division
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Limit the number of greetings to one per thread
    if (idx == 0) {
        printf("Hello CUDA world \n");
    }
}