#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_transpose_k1(float* input, float* output, const int nx, const int ny)
{
    // Calculate row and column indices
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Only execute within bounds
    if (row < ny && col < nx) {
        // Perform transpose by swapping row and column indices
        output[col * ny + row] = input[row * nx + col];
    }
}