#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_details()
{
    // Group related variables into a single printf to reduce overhead
    printf("blockIdx: (%d, %d, %d), blockDim: (%d, %d), gridDim: (%d, %d)\n",
           blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, gridDim.x, gridDim.y);
}