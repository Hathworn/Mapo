#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction_unrolling_blocks4(int * input, int * temp, int size)
{
    int tid = threadIdx.x;
    int BLOCK_OFFSET = blockIdx.x * blockDim.x * 4;
    int index = BLOCK_OFFSET + tid;
    int * i_data = input + BLOCK_OFFSET;

    // Unroll and partially compute the sum if within bounds
    int sum = 0;
    if ((index + 3 * blockDim.x) < size)
    {
        sum = input[index] + input[index + blockDim.x]
            + input[index + 2 * blockDim.x] + input[index + 3 * blockDim.x];
    }
    i_data[tid] = sum;

    __syncthreads();

    // Perform reduction in shared memory
    for (int offset = blockDim.x / 2; offset > 0; offset /= 2)
    {
        if (tid < offset)
        {
            i_data[tid] += i_data[tid + offset];
        }
        __syncthreads();
    }

    // Write result for this block to temp
    if (tid == 0)
    {
        temp[blockIdx.x] = i_data[0];
    }
}