#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction_interleaved_unrolling_blocks8_1(int * input, int * temp, int size)
{
    int tid = threadIdx.x;
    
    // element index for this thread
    int index = blockDim.x * blockIdx.x * 8 + threadIdx.x;
    
    // local data pointer
    int * i_data = input + blockDim.x * blockIdx.x * 8;

    // Unrolling optimization: combine multiple load operations
    if ((index + 7 * blockDim.x) < size)
    {
        int sum = 0;
        sum += input[index];
        sum += input[index + blockDim.x];
        sum += input[index + 2 * blockDim.x];
        sum += input[index + 3 * blockDim.x];
        sum += input[index + 4 * blockDim.x];
        sum += input[index + 5 * blockDim.x];
        sum += input[index + 6 * blockDim.x];
        sum += input[index + 7 * blockDim.x];

        input[index] = sum;
    }
    
    __syncthreads();
    
    // Parallel reduction using interleaved addressing
    for (int offset = blockDim.x / 2; offset > 0; offset /= 2)
    {
        if (tid < offset)
        {
            i_data[tid] += i_data[tid + offset];
        }
        __syncthreads();
    }
    
    if (tid == 0)
    {
        temp[blockIdx.x] = i_data[0];
    }
}