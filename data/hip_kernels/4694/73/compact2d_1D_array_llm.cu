#include "hip/hip_runtime.h"
```cuda
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compact2d_1D_array(int * input, int * output, int * output_column_index_array, int * output_row_index_array, int * prev_output_index_array, int* auxiliry_array, int array_size)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;

    // Early exit for out-of-bound gid
    if (gid >= array_size) return;

    // Perform operation starting from gid = 1
    if (gid > 0)
    {
        // Process elements where previous index differs
        if (prev_output_index_array[gid] != prev_output_index_array[gid - 1])
        {
            // Store the previous input in the current output position
            output[prev_output_index_array[gid]] = input[gid - 1];
            output_column_index_array[prev_output_index_array[gid]] = (gid - 1) % blockDim.x;
        }
        
        int colum_index = gid / (blockDim.x - 1); // Correct typo: column_index
        int condition = gid % (blockDim.x - 1);

        // Check for row index updates
        if (condition == 0)
        {
            if (gid == 0)
            {
                output_row_index_array[0] = 0;
            }
            else
            {
                output_row_index_array[colum_index] = prev_output_index_array[gid];
            }
        }
    }
}