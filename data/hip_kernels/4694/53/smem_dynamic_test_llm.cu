#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smem_dynamic_test(int *in, int *out, int size) 
{
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    extern __shared__ int smem[];

    // Check gid is within valid range to avoid redundant check
    if (gid >= size) return; 

    // Utilize shared memory; coalesced access
    smem[tid] = in[gid];
    __syncthreads(); // Ensure all threads have written to shared memory before reading

    out[gid] = smem[tid];
}