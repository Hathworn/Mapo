#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_smem_pad(int * in, int* out, int nx, int ny)
{
    __shared__ int tile[BDIMY][BDIMX + IPAD];

    int ix = blockDim.x * blockIdx.x + threadIdx.x;
    int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        int in_index = iy * nx + ix;

        // Load input into shared memory
        tile[threadIdx.y][threadIdx.x] = in[in_index];
        __syncthreads();

        // Calculate transposed indices
        int out_ix = blockIdx.y * blockDim.y + threadIdx.x;
        int out_iy = blockIdx.x * blockDim.x + threadIdx.y;

        if (out_ix < ny && out_iy < nx) // Ensure indices are within bounds
        {
            int out_index = out_iy * ny + out_ix;
            out[out_index] = tile[threadIdx.x][threadIdx.y];
        }
    }
}