#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_unroll4_row(int *mat, int *transpose, int nx, int ny) {
    // Calculate row and column index
    int ix = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate linear index for input and output matrices
    int ti = iy * nx + ix;
    int to = ix * ny + iy;

    // Boundary check to ensure threads within matrix dimensions
    if (ix + 3 * blockDim.x < nx && iy < ny) {
        // Unroll the loop manually for better performance
        transpose[to]                     = mat[ti];
        transpose[to + ny * blockDim.x]   = mat[ti + blockDim.x];
        transpose[to + ny * 2 * blockDim.x] = mat[ti + 2 * blockDim.x];
        transpose[to + ny * 3 * blockDim.x] = mat[ti + 3 * blockDim.x];
    }
}