#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_threadIds()
{
    // Use block-wide printf for collective thread coordination
    printf("Block %d, threadIdx.x : %d, threadIdx.y : %d, threadIdx.z : %d \n",
           blockIdx.x, threadIdx.x, threadIdx.y, threadIdx.z);
}