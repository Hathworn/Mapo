#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction_kernel_complete_unrolling8_1(int * input, int * temp, int size)
{
    int tid = threadIdx.x;
    int index = blockDim.x * blockIdx.x * 8 + threadIdx.x;
    int * i_data = input + blockDim.x * blockIdx.x * 8;
    int sum = 0;

    // Check bounds and perform reduction inline
    if ((index + 7 * blockDim.x) < size) {
        for (int offset = 0; offset < 8; ++offset) {
            sum += input[index + offset * blockDim.x];
        }
        input[index] = sum;
    }
    __syncthreads();

    // Complete unrolling explicitly without branching
    if (blockDim.x >= 1024 && tid < 512) i_data[tid] += i_data[tid + 512];
    __syncthreads();
    if (blockDim.x >= 512 && tid < 256) i_data[tid] += i_data[tid + 256];
    __syncthreads();
    if (blockDim.x >= 256 && tid < 128) i_data[tid] += i_data[tid + 128];
    __syncthreads();
    if (blockDim.x >= 128 && tid < 64) i_data[tid] += i_data[tid + 64];
    __syncthreads();

    // Warp unrolling using volatile to prevent reordering
    if (tid < 32) {
        volatile int * vsmem = i_data;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    if (tid == 0) {
        temp[blockIdx.x] = i_data[0];
    }
}