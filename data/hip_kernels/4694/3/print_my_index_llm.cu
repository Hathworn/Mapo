#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_my_index()
{
    // Calculate global thread index for better understanding and usage
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Global ID: %d, Thread ID: %d, Block ID: %d\n", globalId, threadIdx.x, blockIdx.x);
}