#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_smem_pad_unrolling(int * in, int* out, int nx, int ny)
{
    __shared__ int tile[BDIMY * (2 * BDIMX + IPAD)];

    // Calculate input positions
    int ix = 2 * blockDim.x * blockIdx.x + threadIdx.x;
    int iy = blockDim.y * blockIdx.y + threadIdx.y;
    int in_index = iy * nx + ix;

    // Calculate shared memory index
    int _1d_index = threadIdx.y * blockDim.x + threadIdx.x;
    int i_row = _1d_index / blockDim.x;
    int i_col = _1d_index % blockDim.x;

    // Calculate output coordinates
    int out_ix = blockIdx.y * blockDim.y + i_col;
    int out_iy = 2 * blockIdx.x * blockDim.x + i_row;
    int out_index = out_iy * ny + out_ix;
    
    // Check bounds and perform transpose
    if (ix < nx && iy < ny)
    {
        int row_idx = threadIdx.y * (2 * blockDim.x + IPAD) + threadIdx.x;
        
        // Load input into shared memory
        tile[row_idx] = in[in_index];
        if (ix + BDIMX < nx) // Ensure we don't read out of bounds
            tile[row_idx + BDIMX] = in[in_index + BDIMX];
        
        // Wait for all threads to complete loading
        __syncthreads();
        
        int col_idx = i_col * (2 * blockDim.x + IPAD) + i_row;
        
        // Store to output from shared memory
        out[out_index] = tile[col_idx];
        if (out_iy + BDIMX < ny) // Ensure we don't write out of bounds
            out[out_index + ny * BDIMX] = tile[col_idx + BDIMX];
    }
}