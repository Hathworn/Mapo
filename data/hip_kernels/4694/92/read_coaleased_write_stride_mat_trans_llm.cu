#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Global kernel function to optimize matrix transposition
__global__ void read_coaleased_write_stride_mat_trans(float* input, float* output, const int nx, const int ny)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check the bounds condition
    if (ix < nx && iy < ny)
    {
        // Coalesced read and write pattern
        output[iy * nx + ix] = input[ix * ny + iy];
    }
}