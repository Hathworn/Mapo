#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void check_index()
{
    // Use single printf call for clarity and performance
    printf("ThreadIdx: (%d,%d,%d) BlockIdx: (%d,%d,%d) BlockDim: (%d,%d,%d) GridDim: (%d,%d,%d)\n",
           threadIdx.x, threadIdx.y, threadIdx.z,
           blockIdx.x, blockIdx.y, blockIdx.z,
           blockDim.x, blockDim.y, blockDim.z,
           gridDim.x, gridDim.y, gridDim.z);
}