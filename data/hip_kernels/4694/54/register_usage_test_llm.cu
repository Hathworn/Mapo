#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void register_usage_test(int * results, int size)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;

    // Combined redundant variables into one calculation
    int x4 = (3465 * 2) + (1768 * 2) + (453 * 2);

    if (gid < size)
    {
        results[gid] = x4;
    }
}