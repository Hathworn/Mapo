#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void read_stride_write_coaleased_mat_trans(float* input, float* output, const int nx, const int ny)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        // Utilize shared memory for coalesced memory access
        __shared__ float tile[BLOCK_SIZE][BLOCK_SIZE + 1];
        
        // Read into shared memory in a coalesced manner
        tile[threadIdx.y][threadIdx.x] = input[ix * ny + iy];
        
        __syncthreads();
        
        // Write from shared memory to global memory in a coalesced manner
        output[iy * nx + ix] = tile[threadIdx.y][threadIdx.x];
    }
}