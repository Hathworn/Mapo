#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_details_of_warps() 
{
    // Compute global thread ID
    int gid = blockIdx.y * gridDim.x * blockDim.x
            + blockIdx.x * blockDim.x + threadIdx.x;

    // Efficiently calculate warp ID within block
    int warp_id = threadIdx.x >> 5;  // Equivalent to threadIdx.x / 32 using bitshift

    int gbid = blockIdx.y * gridDim.x + blockIdx.x;

    // Reduce printf frequency by only letting one thread per warp print
    if (threadIdx.x % 32 == 0) {
        printf("tid : %d, bid.x : %d, bid.y : %d, gid : %d, warp_id : %d, gbid : %d \n",
               threadIdx.x, blockIdx.x, blockIdx.y, gid, warp_id, gbid);
    }
}