#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void misaligned_write_test(float* a, float* b, float *c, int size, int offset)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int k = gid + offset;

    if (k < size)
    {
        float val_a = a[gid]; // Preload value from a[gid]
        float val_b = b[gid]; // Preload value from b[gid]
        c[k] = val_a + val_b; // Use preloaded values
    }
}