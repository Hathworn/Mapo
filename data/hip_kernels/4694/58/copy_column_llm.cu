#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_column(int *mat, int *transpose, int nx, int ny)
{
    // Calculate single unique index for 2D grid
    int idx = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx >= ny) return;  // Early exit if out of bounds on y-dimension

    int idy = blockIdx.x * blockDim.x + threadIdx.x;
    if (idy >= nx) return;  // Early exit if out of bounds on x-dimension

    // Efficient memory access with 1D index
    transpose[idy * ny + idx] = mat[idy * ny + idx];
}