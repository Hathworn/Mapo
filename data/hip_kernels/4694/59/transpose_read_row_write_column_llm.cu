#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_read_row_write_column(int * mat, int * transpose, int nx, int ny)
{
    // Calculate global id
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure indices are within bounds
    if (ix < nx && iy < ny)
    {
        // Coalesced memory access
        int index_in = iy * nx + ix;
        int index_out = ix * ny + iy;
        transpose[index_out] = mat[index_in];
    }
}