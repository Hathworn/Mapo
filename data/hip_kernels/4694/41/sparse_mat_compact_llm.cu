#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sparse_mat_compact(int * input, int * output, int * output_index_array, int array_size)
{
    int gid = blockDim.x * blockIdx.x + threadIdx.x;

    // Improved memory access by including gid == 0, minimizing conditional checks
    if (gid < array_size)
    {
        int current_index = output_index_array[gid];
        if (gid == 0 || current_index != output_index_array[gid - 1])
        {
            // Directly store input value to avoid additional look-up
            output[current_index] = gid == 0 ? input[gid] : input[gid - 1];
        }
    }
}