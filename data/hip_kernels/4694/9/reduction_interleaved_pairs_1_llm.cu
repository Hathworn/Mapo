#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction_interleaved_pairs_1(int *int_array, int *temp_array, int size) {
    int tid = threadIdx.x;
    int gid = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure gid is within bounds
    if (gid >= size) return;

    // Optimized reduction kernel
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (tid < offset && gid + offset < size) { // Prevent accessing out of bounds
            int_array[gid] += int_array[gid + offset];
        }
        __syncthreads();
    }

    // Store the result of this block in global memory
    if (tid == 0) {
        temp_array[blockIdx.x] = int_array[gid];
    }
}