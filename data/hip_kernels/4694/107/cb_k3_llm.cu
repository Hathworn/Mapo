#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel launch by reducing blocks if possible
__global__ void cb_k3()
{
    // Use early exit to avoid unnecessary calculations for all threads
    if (blockIdx.x != 0 || threadIdx.x != 0) return;

    // Use printf sparingly in kernel functions
    printf("This is a test 3 \n");
}