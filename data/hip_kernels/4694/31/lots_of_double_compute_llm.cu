#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lots_of_double_compute(double *inputs, int N, size_t niters, double *outputs)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t nthreads = gridDim.x * blockDim.x;

    // Loop optimization: unrolling the iteration loop
    for (; tid < N; tid += nthreads)
    {
        size_t iter;
        double val = inputs[tid];

        for (iter = 0; iter < niters; iter += 6) // Unroll by a factor of 6
        {
            val += 5.0;
            val -= 101.0;
            val /= 3.0;
            val += 102.0;
            val += 1.07;
            val -= 103.0;
            val /= 1.037;
            val += 104.0;
            val += 3.00;
            val -= 105.0;
            val /= 0.22;
            val += 106.0;
        }

        // If niters is not a multiple of 6, handle remaining iterations
        int remaining = niters % 6;
        for (iter = 0; iter < remaining; iter++)
        {
            val = (val + 5.0) - 101.0;
            val = (val / 3.0) + 102.0;
            val = (val + 1.07) - 103.0;
            val = (val / 1.037) + 104.0;
            val = (val + 3.00) - 105.0;
            val = (val / 0.22) + 106.0;
        }

        outputs[tid] = val;
    }
}