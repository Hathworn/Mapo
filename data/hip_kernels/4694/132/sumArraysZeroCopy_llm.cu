#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysZeroCopy(int *A, int *B, int *C, const int N)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilize shared memory to minimize global memory access
    extern __shared__ int sharedData[];
    int *sA = sharedData;
    int *sB = sharedData + blockDim.x;

    // Load data into shared memory
    if (i < N) {
        sA[threadIdx.x] = A[i];
        sB[threadIdx.x] = B[i];
    }
    __syncthreads();

    // Perform computation using shared memory
    if (i < N) {
        C[i] = sA[threadIdx.x] + sB[threadIdx.x];
    }
}