#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compact_1D_array(int *input, int *output, int *output_index_array, int array_size) {
    int gid = blockDim.x * blockIdx.x + threadIdx.x;

    // Improved memory access pattern
    if (gid < array_size) {
        int current_index = output_index_array[gid];
        if (gid > 0) {
            int previous_index = output_index_array[gid - 1];
            if (current_index != previous_index) {
                output[current_index] = input[gid - 1];
            }
        } else if (current_index == 0) {
            // Handle the first element case
            output[current_index] = input[gid];
        }
    }
}