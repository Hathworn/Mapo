#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BDIMX 16 // Define block dimension X
#define BDIMY 16 // Define block dimension Y

__global__ void transpose_smem(int *in, int *out, int nx, int ny)
{
    __shared__ int tile[BDIMY][BDIMX];

    // Calculate input indices
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate transposed indices
    unsigned int transposed_ix = blockIdx.y * blockDim.y + threadIdx.x;
    unsigned int transposed_iy = blockIdx.x * blockDim.x + threadIdx.y;

    // Ensure indices are within bounds before accessing
    if (ix < nx && iy < ny)
    {
        // Load into shared memory
        tile[threadIdx.y][threadIdx.x] = in[iy * nx + ix];
    }

    // Synchronize threads to ensure all have loaded
    __syncthreads();

    // Output results to global memory if within bounds
    if (transposed_ix < ny && transposed_iy < nx)
    {
        out[transposed_iy * ny + transposed_ix] = tile[threadIdx.x][threadIdx.y];
    }
}