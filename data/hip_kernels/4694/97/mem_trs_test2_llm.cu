#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mem_trs_test2(int * input, int size)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Limit work to threads within grid size
    if (gid < size) {
        // Use threadfence to ensure memory visibility
        __threadfence();

        // Efficient conditional logging
        printf("tid : %d , gid : %d, value : %d \n", threadIdx.x, gid, input[gid]);
    }
}