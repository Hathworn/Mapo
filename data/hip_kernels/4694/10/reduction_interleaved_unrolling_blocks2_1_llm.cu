#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction_interleaved_unrolling_blocks2_1(int * input, int * temp, int size)
{
    extern __shared__ int s_data[];  // Use shared memory to optimize memory access
    int tid = threadIdx.x;
    
    // Start index for this thread
    int index = blockDim.x * blockIdx.x * 2 + threadIdx.x;
    
    // Load elements into shared memory
    if (index < size) {
        s_data[tid] = input[index];
        if (index + blockDim.x < size) {
            s_data[tid] += input[index + blockDim.x];
        }
    } else {
        s_data[tid] = 0;
    }
    
    __syncthreads();

    // Unroll and reduce within shared memory
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (tid < offset) {
            s_data[tid] += s_data[tid + offset];
        }
        __syncthreads();
    }
    
    // Store result from block to global memory
    if (tid == 0) {
        temp[blockIdx.x] = s_data[0];
    }
}