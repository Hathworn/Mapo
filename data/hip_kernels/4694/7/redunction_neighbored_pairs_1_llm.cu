#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void redunction_neighbored_pairs_1(int * input, int * temp, int size)
{
    int tid = threadIdx.x;
    int gid = blockDim.x * blockIdx.x + threadIdx.x;

    // Return immediately if gid is out of bounds
    if (gid >= size)
        return;

    // Using more efficient loop to reduce
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1)
    {
        if (tid < offset)
        {
            input[gid] += input[gid + offset];
        }

        __syncthreads(); // Ensure all threads have completed addition
    }

    // Write block result to temp only if tid is 0
    if (tid == 0)
    {
        temp[blockIdx.x] = input[gid];
    }
}