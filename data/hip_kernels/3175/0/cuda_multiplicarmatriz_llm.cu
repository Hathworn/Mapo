#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TAM_BLOCO 16

__global__ void cuda_multiplicarmatriz(float* M, float* N, float* R, int tamM, int tamN) {

    // Block and thread indices
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Compute starting points
    int mComeco = tamM * TAM_BLOCO * by;
    int mFim   = mComeco + tamM - 1;
    int mPasso  = TAM_BLOCO;
    int nComeco = TAM_BLOCO * bx;
    int nPasso  = TAM_BLOCO * tamN;

    float rRes = 0;

    // Loop through submatrices
    for (int m = mComeco, n = nComeco; m <= mFim; m += mPasso, n += nPasso) {
        
        // Shared memory allocation
        __shared__ float Msub[TAM_BLOCO][TAM_BLOCO];
        __shared__ float Nsub[TAM_BLOCO][TAM_BLOCO];

        // Load matrices into shared memory
        Msub[ty][tx] = M[m + tamM * ty + tx];
        Nsub[ty][tx] = N[n + tamN * ty + tx];
        __syncthreads();

        // Matrix multiplication loop
        #pragma unroll // Unroll loop for better performance
        for (int i = 0; i < TAM_BLOCO; ++i)
            rRes += Msub[ty][i] * Nsub[i][tx];
        __syncthreads();
    }

    // Write to global memory
    int r = tamN * TAM_BLOCO * by + TAM_BLOCO * bx;
    R[r + tamN * ty + tx] = rRes;
}