#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel with grid-stride loop for better scalability
__global__ void cuda_olamundo() {
    // Calculate unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a grid-stride loop to handle more threads if needed
    for (int i = idx; i < 1; i += gridDim.x * blockDim.x) {
        printf("Ola Mundo direto da GPU!\n");
    }
}