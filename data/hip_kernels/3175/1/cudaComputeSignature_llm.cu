#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaComputeSignature(double* hyperplanes, double* v, int* dimensions, bool* sig, long* hyperp_length) {
    long tid = threadIdx.x + blockDim.x * blockIdx.x;

    if (tid < *hyperp_length) {
        int d_dimensions = *dimensions;
        long pos = tid * d_dimensions;
        double sum = 0.0;

        // Utilize shared memory for better memory access pattern
        extern __shared__ double shared_v[];
        
        // Each thread loads a part of the vector v into shared memory
        if (threadIdx.x < d_dimensions) {
            shared_v[threadIdx.x] = v[threadIdx.x];
        }
        __syncthreads();

        // Compute dot product using shared memory
        for (int i = 0; i < d_dimensions; i++) {
            sum += hyperplanes[i + pos] * shared_v[i];
        }

        sig[tid] = (sum >= 0);
    }
}