#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kLogregCost kernel function
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Optimize probability computation using a single comparison.
        int numMax = (labelp == maxp) ? 1 : 0;
        for (int i = 0; i < numOut; i++) {
            // Accumulate the count of maximum probabilities
            numMax += probs[i * numCases + tx] == maxp;
        }
        correctProbs[tx] = (labelp == maxp) ? (1.0f / float(numMax)) : 0.0f;
    }
}