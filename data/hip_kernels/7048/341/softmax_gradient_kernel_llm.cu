#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    float tmp = 0;

    // Loop unrolling to enhance performance.
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }

    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Use atomicAdd for efficient reduction.
    if (idx == 0) {
        tmp = 0;
        for (int i = 0; i < blockDim.x; ++i) {
            tmp += reduction_buffer[i];
        }
        reduction_buffer[0] = tmp;
    }
    __syncthreads();

    // Compute gradient efficiently.
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}