#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Using memory coalescing for better global memory access
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x; 
    /* Each thread gets same seed, a different sequence number, no offset */
    hiprand_init(seed, tidx, 0, &state[tidx]);
}