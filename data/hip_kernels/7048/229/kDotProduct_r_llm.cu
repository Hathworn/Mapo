#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define WARP_SIZE 32

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Preload loop into registers for reduction
    if (eidx < numElements) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            sum += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Unrolled loop to reduce within shared memory using warp shuffle
    if (threadIdx.x < WARP_SIZE) {
        volatile float* mysh = &shmem[threadIdx.x];
        mysh[0] += mysh[32];
        mysh[0] += mysh[16];
        mysh[0] += mysh[8];
        mysh[0] += mysh[4];
        mysh[0] += mysh[2];
        mysh[0] += mysh[1];
        if (threadIdx.x == 0) {
            target[blockIdx.x] = mysh[0];
        }
    }
}