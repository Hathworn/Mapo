#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void copy_sort_int(const int *orig, const unsigned int *sort_idx, const unsigned int nitems, int *sorted) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < nitems) { // Ensure index is within bounds
        sorted[sort_idx[i]] = orig[i]; // Copy and sort element
    }
}