#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void average_snips_v2(const double *Params, const int *iC, const int *call, const int *id, const float *uproj, const float *cmax, float *WU) {
    // Optimize: Use shared memory for frequently accessed params
    extern __shared__ int shared_iC[];
    int Nspikes = (int) Params[0];
    int NrankPC = (int) Params[1];
    int Nchan = (int) Params[7];
    int NchanNear = (int) Params[6];
    
    int bid = blockIdx.x;
    float xsum = 0.0f;

    // Cache iC in shared memory
    if (threadIdx.x < NchanNear * Nchan) {
        shared_iC[threadIdx.x] = iC[threadIdx.x];
    }
    __syncthreads();
    
    // Process spikes for this block
    for (int ind = threadIdx.x; ind < Nspikes; ind += blockDim.x) {
        if (id[ind] == bid) {
            int my_chan = call[ind];
            for (int chanIndex = 0; chanIndex < NchanNear; ++chanIndex) {
                int this_chan = shared_iC[chanIndex + NchanNear * my_chan];
                for (int pcIndex = 0; pcIndex < NrankPC; ++pcIndex) {
                    xsum = uproj[pcIndex + NrankPC * chanIndex + NrankPC * NchanNear * ind];
                    atomicAdd(&WU[pcIndex + NrankPC * this_chan + NrankPC * Nchan * bid], xsum); // Optimize: Use atomicAdd to avoid collisions
                }
            }
        }
    }
}