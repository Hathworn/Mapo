#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bestFilter(const double *Params, const bool *iMatch, const int *Wh, const float *cmax, const float *mus, int *id, float *x) {

    int tid, tind, my_chan, Nspikes, Nfilters, Nthreads, Nchan, Nblocks;
    float max_running = 0.0f;

    Nspikes = static_cast<int>(Params[0]);
    Nfilters = static_cast<int>(Params[2]);
    Nchan = static_cast<int>(Params[7]);
    Nthreads = blockDim.x;
    Nblocks = gridDim.x;

    tid = threadIdx.x;
    int bid = blockIdx.x;

    tind = tid + bid * Nthreads;

    while (tind < Nspikes) {
        // Use mus[tind] squared value for comparison directly
        max_running = mus[tind] * mus[tind];
        id[tind] = 0;
        my_chan = Wh[tind];
        for (int ind = 0; ind < Nfilters; ++ind) {
            if (iMatch[my_chan + ind * Nchan]) {
                int cmax_index = tind + ind * Nspikes;
                if (cmax[cmax_index] < max_running) {
                    id[tind] = ind;
                    max_running = cmax[cmax_index];
                }
            }
        }
        x[tind] = max_running;
        tind += Nblocks * Nthreads;
    }
}