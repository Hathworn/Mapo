#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void average_snips_v3(const double *Params, const int *ioff, const int *id, const float *uproj, const float *cmax, float *bigArray) {
    // Load constant parameters into registers 
    int Nspikes = (int) Params[0];
    int Nfeatures = (int) Params[1];
    int NfeatW = (int) Params[4];
    
    // Calculate thread and block index
    int tid = threadIdx.x; // feature index
    int bid = blockIdx.x;  // filter index

    // Use shared memory to cache data for better performance
    __shared__ float sharedUproj[Nfeatures];

    for (int ind = 0; ind < Nspikes; ind++) {
        // Load uproj portion into shared memory
        if (tid < Nfeatures) {
            sharedUproj[tid] = uproj[tid + Nfeatures * ind];
        }
        __syncthreads();

        if (id[ind] == bid) {
            // Access shared memory to reduce global memory reads
            float xsum = sharedUproj[tid];
            atomicAdd(&bigArray[ioff[ind] + tid + tid * NfeatW + Nfeatures * NfeatW * bid], xsum);
        }
        __syncthreads(); // Ensure all threads completed the operation before next iteration
    }
}