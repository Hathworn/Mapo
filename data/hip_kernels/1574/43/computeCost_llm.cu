#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeCost(const double *Params, const float *uproj, const float *mu, const float *W, const bool *match, const int *iC, const int *call, float *cmax){

    int NrankPC, j, NchanNear, tid, bid, Nspikes, Nthreads, k, my_chan, this_chan, Nchan;
    float xsum, Ci, lam;

    Nspikes    = (int) Params[0];
    NrankPC    = (int) Params[1];
    Nthreads   = blockDim.x;
    lam        = (float) Params[5];
    NchanNear  = (int) Params[6];
    Nchan      = (int) Params[7];

    tid        = threadIdx.x + blockIdx.x * blockDim.x; // Use unique thread index for 1D grid
    bid        = blockIdx.x;

    // Iterate over spikes with stride equal to the total number of threads
    while (tid < Nspikes) {
        my_chan = call[tid];
        if (match[my_chan + bid * Nchan]) {
            xsum = 0.0f;
            // Optimize nested loop by unrolling if possible
            for (k = 0; k < NchanNear; k++) {
                for (j = 0; j < NrankPC; j++) {
                    this_chan = iC[k + my_chan * NchanNear];
                    xsum += uproj[j + NrankPC * k + NrankPC * NchanNear * tid] *
                            W[j + NrankPC * this_chan + NrankPC * Nchan * bid];
                }
            }
            Ci = fmaxf(0.0f, xsum) + lam / mu[bid];

            // Directly compute the final result
            float mu_bid_squared = mu[bid] * mu[bid];
            cmax[tid + bid * Nspikes] = (Ci * Ci / (1.0f + lam / mu_bid_squared)) - lam;
        }
        tid += blockDim.x * gridDim.x; // Ensure all spikes are covered by block and grid
    }
}