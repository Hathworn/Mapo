#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void average_snips(const double *Params, const int *iC, const int *call, const int *id, const float *uproj, const float *cmax, float *WU) {
    // Cache parameters in local variables for constant memory access
    int Nspikes = (int)Params[0];
    int NrankPC = (int)Params[1];
    int Nchan = (int)Params[7];
    int NchanNear = (int)Params[6];

    // Use local variables for thread and block indices
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int bid = blockIdx.x;

    float xsum;

    // Use shared memory to reduce global memory accesses
    __shared__ float shared_uproj[1024]; // Adjust size as needed

    for (int ind = tidx; ind < Nspikes; ind += blockDim.x) {
        if (id[ind] == bid) {
            int my_chan = call[ind];
            int this_chan = iC[tidy + NchanNear * my_chan];

            // Preload data into shared memory for coalesced access
            shared_uproj[tidx] = uproj[tidx + NrankPC * tidy + NrankPC * NchanNear * ind];
            __syncthreads(); // Ensure all threads have loaded the data

            xsum = shared_uproj[tidx];
            atomicAdd(&WU[tidx + NrankPC * this_chan + NrankPC * Nchan * bid], xsum); // Use atomicAdd to prevent collisions
        }
    }
}