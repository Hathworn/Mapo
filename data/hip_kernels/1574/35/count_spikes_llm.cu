#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void count_spikes(const double *Params, const int *id, int *nsp) {
    int Nspikes = (int) Params[0];
    int Nfilters = (int) Params[2];
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int Nthreads = blockDim.x;
    int Nblocks = gridDim.x;
    
    __shared__ int local_nsp[1024];  // Use shared memory for partial sums
    local_nsp[tid] = 0;
    __syncthreads();

    int tind = tid + Nthreads * bid;
    while (tind < Nfilters) {
        for (int ind = 0; ind < Nspikes; ind++) {
            if (id[ind] == tind) {
                atomicAdd(&local_nsp[tid], 1);  // Accumulate using atomic add
            }
        }
        tind += Nthreads * Nblocks;
    }

    __syncthreads();

    // Reduce shared memory results to global memory
    if (tid == 0) {
        int sum = 0;
        for (int i = 0; i < Nthreads; i++) {
            sum += local_nsp[i];
        }
        atomicAdd(&nsp[bid], sum);
    }
}