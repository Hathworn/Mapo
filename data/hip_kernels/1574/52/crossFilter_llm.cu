#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void crossFilter(const double *Params, const float *W1, const float *W2, const float *UtU, float *WtW) {
    // Utilize shared memory effectively and manage thread indices
    __shared__ float shW1[nblock * 81], shW2[nblock * 81];

    float x;
    int nt0, tidx, tidy, bidx, bidy, Nfilt, t, tid1, tid2;

    tidx = threadIdx.x;
    tidy = threadIdx.y;
    bidx = blockIdx.x;
    bidy = blockIdx.y;

    Nfilt = (int)Params[1];
    nt0 = (int)Params[9];

    tid1 = tidx + bidx * nblock;
    tid2 = tidy + bidy * nblock;

    // Load W1 into shared memory using coalesced access
    if (tid2 < Nfilt) {
        #pragma unroll
        for (int i = tidx; i < nt0; i += nblock) {
            shW1[i + tidy * nt0] = W1[i + tid2 * nt0];
        }
    }

    // Reset tidx for loading W2
    tidx = threadIdx.x;
    tid2 = tidy + bidy * nblock;

    // Load W2 into shared memory using coalesced access
    if (tid2 < Nfilt) {
        #pragma unroll
        for (int i = tidx; i < nt0; i += nblock) {
            shW2[i + tidy * nt0] = W2[i + tid2 * nt0];
        }
    }

    __syncthreads();

    // Compute cross filter results with shared memory and using loops effectively
    if (tid2 < Nfilt && tid1 < Nfilt) {
        for (i = 0; i < 2 * nt0 - 1; i++) {
            x = 0.0f;

            // Optimize computation using two separate loops
            if (i < nt0) {
                #pragma unroll
                for (t = 0; t <= i; t++) {
                    x += shW1[t + nt0 * tidx] * shW2[t + (nt0 - i - 1) + nt0 * tidy];
                }
            } else {
                #pragma unroll
                for (t = i - nt0 + 1; t < nt0; t++) {
                    x += shW1[t + nt0 * tidx] * shW2[t + (nt0 - i - 1) + nt0 * tidy];
                }
            }

            WtW[tid1 + tid2 * Nfilt + i * Nfilt * Nfilt] = x * UtU[tid1 + tid2 * Nfilt];
        }
    }
}