#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void count_spikes(const double *Params, const int *id, int *nsp, const float *x, float *V) {
    // Use shared memory to optimize memory access
    extern __shared__ int shared_nsp[];
    
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int NthreadsMe = blockDim.x;
    int Nblocks = gridDim.x;
    
    int Nspikes = (int) Params[0];
    int Nfilters = (int) Params[2];
    
    int tind = tid + NthreadsMe * bid;
    
    // Initialize shared memory
    if (tid < Nfilters) {
        shared_nsp[tid] = 0;
    }
    __syncthreads();
    
    while (tind < Nfilters) {
        float sum_V = 0.0f;

        for (int ind = 0; ind < Nspikes; ind++) {
            if (id[ind] == tind) {
                atomicAdd(&shared_nsp[tid], 1);  // Use atomic operations for thread safety
                sum_V += x[tind];
            }
        }
        
        // Accumulate result in global memory
        atomicAdd(&nsp[tind], shared_nsp[tid]);
        V[tind] += sum_V;
        
        // Normalize the voltage V
        V[tind] = V[tind] / (.001f + (float) nsp[tind]);

        tind += NthreadsMe * Nblocks;
    }
}