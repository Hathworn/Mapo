#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void max1D(const double *Params, const float *data, float *conv_sig) {
    // Use dynamic shared memory for better flexibility
    extern __shared__ float sdata[];

    float y, spkTh;
    int tid, tid0, bid, NT, nt0;

    NT = (int) Params[0];
    nt0 = (int) Params[3];
    spkTh = (float) Params[5];
    tid = threadIdx.x;
    bid = blockIdx.x;

    tid0 = 0;
    while (tid0 < NT - Nthreads - nt0 + 1) {
        if (tid < nt0) {
            // Coalesced memory access for better performance
            sdata[tid] = data[tid0 + tid + NT * bid];
        }
        sdata[tid + nt0] = data[nt0 + tid0 + tid + NT * bid];

        __syncthreads();

        y = 0.0f;
        // Loop unrolling for improved throughput
        #pragma unroll 4
        for (i = 0; i < nt0; i++) {
            // Use FMA for potential further optimization
            y = max(y, sdata[tid + i]);
        }

        if (y > spkTh) {
            // Coalesced global memory write
            conv_sig[tid0 + tid + NT * bid] = y;
        }

        tid0 += Nthreads;
        __syncthreads();
    }
}