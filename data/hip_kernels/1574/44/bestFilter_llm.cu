#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bestFilter(const double *Params, const bool *match, const int *iC, const int *call, const float *cmax, int *id, float *cx) {
    int Nspikes = (int) Params[0];
    int Nfilters = (int) Params[2];
    int Nchan = (int) Params[7];

    int tid = threadIdx.x;
    int bid = blockIdx.x;

    int tind = tid + bid * blockDim.x * gridDim.x;  // Optimize thread index calculation for coalesced access

    while (tind < Nspikes) {
        float max_running = -1.0f;  // Initialize to -1.0 for valid max checks
        int my_chan = call[tind];  // Cache call for current spike
        id[tind] = 0;  // Assigning default best filter id

        for (int ind = 0; ind < Nfilters; ind++) {
            int match_idx = my_chan + ind * Nchan;
            if (match[match_idx]) {
                int cmax_idx = tind + ind * Nspikes;
                float current_cmax = cmax[cmax_idx];  // Cache current cmax value
                if (current_cmax > max_running) {  // Compare with cached max value
                    id[tind] = ind;  // Update best filter id
                    max_running = current_cmax;  // Update running max
                }
            }
        }

        cx[tind] = max_running;  // Store the max cmax value for the current spike

        tind += blockDim.x * gridDim.x;  // Increment to next element step size
    }
}