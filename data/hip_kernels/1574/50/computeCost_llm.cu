#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeCost(const double *Params, const float *Ws, const float *mus, const float *W, const float *mu, const bool *iMatch, const int *iC, const int *Wh, float *cmax) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;  // Unique global thread index
    int stride = blockDim.x * gridDim.x;  // Stride determined by total threads

    int Nspikes = (int) Params[0];  // Number of spikes/comparisons
    int NrankPC = (int) Params[1];
    int Nchan = (int) Params[7];
    int NchanNear = (int) Params[6];

    while (tid < Nspikes) {
        int my_chan = Wh[tid];
        if (iMatch[my_chan + blockIdx.x * Nchan]) {  // Check channel
            float xsum = 0.0f;
            for (int k = 0; k < NchanNear; k++) {
                int this_chan = iC[k + NchanNear * my_chan];
                for (int j = 0; j < NrankPC; j++) {
                    // Accumulate product sum
                    xsum += Ws[j + NrankPC * k + NrankPC * NchanNear * tid] * W[j + NrankPC * this_chan + NrankPC * Nchan * blockIdx.x];
                }
            }
            float Ci = mu[blockIdx.x] * mu[blockIdx.x] + mus[tid] * mus[tid] - 2 * mus[tid] * mu[blockIdx.x] * xsum;
            cmax[tid + blockIdx.x * Nspikes] = Ci;  // Update result
        }
        tid += stride;  // Advance by complete set of threads
    }
}