#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_dWU(const double *Params, const float *bigArray, float *WU) {

    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int Nfilters = (int) Params[2];
    int Nthreads = blockDim.x;
    int Nfeatures = (int) Params[1];
    int Nblocks = gridDim.x;
    int NfeatW = (int) Params[4];

    int nWU = NfeatW * Nfilters;
    int nElem = Nfeatures * NfeatW; // number of elements in each subArray of bigArray

    // Calculate which element we're addressing
    int tind = tid + bid * Nthreads;

    // Use loop unrolling strategy for better performance
    while (tind < nWU) {
        int currFilt = tind / NfeatW; // Use integer division for faster calculation
        int currFW = tind % NfeatW;   // Use modulo for faster calculation

        float sum = 0.0f;

#pragma unroll
        for (int ind = 0; ind < Nfeatures; ind++) {
            int currIndex = currFilt * nElem + ind * NfeatW + currFW; // Calculate index directly
            sum += bigArray[currIndex];
        }

        WU[tind] += sum;
        tind += Nblocks * Nthreads;
    }
}