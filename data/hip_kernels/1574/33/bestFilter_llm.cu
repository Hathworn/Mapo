#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bestFilter(const double *Params, const bool *iW, const float *cmax, int *id) {

    int tind = threadIdx.x + blockIdx.x * blockDim.x; // Direct calculation of tind
    int Nspikes = (int) Params[0];
    int Nfilters = (int) Params[2];
    float Th = (float) Params[7];

    while (tind < Nspikes) {
        float max_running = 0.0f;
        id[tind] = 0;

        for (int ind = 0; ind < Nfilters; ind++) {
            int idx = tind + ind * Nspikes; // Precompute index to avoid recalculation
            if (iW[idx] && cmax[idx] > max_running) { 
                id[tind] = ind;
                max_running = cmax[idx];
            }
        }

        if (max_running < Th * Th) {
            id[tind] = -1; // Assign -1 if below threshold
        }

        tind += gridDim.x * blockDim.x; // Increment by total number of threads
    }
}