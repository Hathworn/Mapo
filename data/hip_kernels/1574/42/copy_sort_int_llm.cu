#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_sort_int(const float *orig, const unsigned int *sort_idx, const unsigned int nitems, float *sorted) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Each thread handles one index
    if (idx < nitems) {
        sorted[sort_idx[idx]] = orig[idx];
    }
}