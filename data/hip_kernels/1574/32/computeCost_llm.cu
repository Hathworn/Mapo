#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeCost(const double *Params, const float *uproj, const float *mu, const float *W, const int *ioff, const bool *iW, float *cmax) {
    // Precompute constant values outside the loop
    int Nspikes = (int) Params[0];
    int Nfeatures = (int) Params[1];
    int NfeatW = (int) Params[4];
    float lam = (float) Params[5];

    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int threadTotal = gridDim.x * blockDim.x;

    while (tid < Nspikes) {
        if (iW[tid + blockIdx.x * Nspikes]) {
            float xsum = 0.0f;
            int tidOffset = Nfeatures * tid;
            int bidOffset = NfeatW * blockIdx.x;
            int wOffset = ioff[tid] + bidOffset;
            for (int k = 0; k < Nfeatures; k++) {
                xsum += uproj[k + tidOffset] * W[k + wOffset];
            }

            // Precompute reusable values and simplify expression
            float Ci = max(0.0f, xsum) + lam / mu[blockIdx.x];
            float mu2 = mu[blockIdx.x] * mu[blockIdx.x];
            cmax[tid + blockIdx.x * Nspikes] = Ci * Ci / (1.0f + lam / mu2) - lam;
        }
        // Step through threads more efficiently
        tid += threadTotal;
    }
}