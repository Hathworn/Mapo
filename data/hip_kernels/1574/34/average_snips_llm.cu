#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void average_snips(const double *Params, const int *ioff, const int *id, const float *uproj, const float *cmax, const int *iList, float *cf, float *WU) {

    int tid = threadIdx.x;
    int bid = blockIdx.x;

    float pm = (float)Params[3];
    int Nspikes = (int)Params[0];
    int Nfeatures = (int)Params[1];
    int NfeatW = (int)Params[4];
    int Nnearest = (int)Params[6];

    int ind, t;

    // Loop through spikes only relevant to current block
    for (ind = tid; ind < Nspikes; ind += blockDim.x) {
        if (id[ind] == bid) {
            float xsum = uproj[tid + Nfeatures * ind];
            int wu_index = tid + ioff[ind] + NfeatW * bid;
            
            // Update WU with weighted sum
            WU[wu_index] = pm * WU[wu_index] + (1 - pm) * xsum;

            // Update cf for top nearest matches
            for (t = 0; t < Nnearest; t++) {
                cf[ind + t * Nspikes] = cmax[ind + Nspikes * iList[t + Nnearest * bid]];
            }
        }
    }
}