#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(unsigned int rows, unsigned int cols, float* ddata, float* vdata, float* results) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    float dp = 0;
    
    // Use shared memory for faster access times compared to global memory
    extern __shared__ float shared_vdata[];
    
    // Load vdata into shared memory
    if (threadIdx.x < cols) {
        shared_vdata[threadIdx.x] = vdata[threadIdx.x];
    }
    __syncthreads();
    
    // Compute the dot product
    for (int i = 0; i < cols; i++) {
        dp += ddata[i * rows + tid] * shared_vdata[i];
    }
    
    // Store the result
    results[tid] = dp;
}