#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Efficiently iterate over the entire range with stride
    for (int32_t i = idx; i < N * gridDim.x; i += blockDim.x * gridDim.x) {
        int matrixRow = i / N;
        int matrixCol = i % N;
        *(output + i) = -row_sum[matrixRow];
    }
}