#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Use thread ID for states instead of single state initialization
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, id, 0, &state[id]);
}