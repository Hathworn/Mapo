#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    int32_t globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride = gridDim.x * blockDim.x;
    // Loop over all elements using a grid-stride loop
    for (int32_t i = globalIdx; i < N; i += stride) {
        output[blockIdx.x * N + i] = -col_sum[i];
    }
}