#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline static float euclid_distance(int numCoords, int tid, int clusterId, float *objects, float *clusters)
{
    float ans = 0.0;
    for (int i = 0; i < numCoords; i++) {
        float diff = objects[tid * numCoords + i] - clusters[clusterId * numCoords + i];
        ans += diff * diff; // Combine subtraction and multiplication
    }
    return ans;
}

__global__ static void find_nearest_cluster(int numCoords, int numObjs, int numClusters, float *objects, float *deviceClusters, int *membership, int *changedmembership)
{
    extern __shared__ float sharedMem[];
    // Shared memory for clusters and objects
    float *sh_Clusters = sharedMem;
    
    // Copy clusters into shared memory
    int idx = threadIdx.x;
    int elemsPerCluster = numCoords;
    while (idx < numClusters * elemsPerCluster) {
        sh_Clusters[idx] = deviceClusters[idx];
        idx += blockDim.x;
    }
    __syncthreads();

    unsigned int tid = threadIdx.x;
    int objectId = blockDim.x * blockIdx.x + tid;

    while (objectId < numObjs) {
        int index = 0;
        float min_dist = euclid_distance(numCoords, tid, 0, &objects[objectId * numCoords], sh_Clusters);

        for (int i = 1; i < numClusters; i++) {
            float dist = euclid_distance(numCoords, tid, i, &objects[objectId * numCoords], sh_Clusters);
            if (dist < min_dist) {
                min_dist = dist;
                index = i;
            }
        }

        if (membership[objectId] != index) {
            changedmembership[objectId] = 1;
            membership[objectId] = index;
        }

        objectId += blockDim.x * gridDim.x;
    }
}