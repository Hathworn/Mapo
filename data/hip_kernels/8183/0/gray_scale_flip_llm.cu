#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/******************************
*STUDENT NAME: DAVID PARKS    *
*PROJECT: 6 - GREY SCALE FLIP *
*DUE DATE: THURS 18/10/18     *
*******************************/

#define PPM_MAGIC_1 'P'
#define PPM_MAGIC_2 '6'
#define BLOCK_SIZE 16;

struct PPM_header {
    int width;
    int height;
    int max_color;
};

struct RGB_8 {
    uint8_t r;
    uint8_t g;
    uint8_t b;
};

__global__ void gray_scale_flip(RGB_8* img, int height, int width)
{
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (row < height && col < width / 2)
    {
        // Calculate indices once to avoid recomputation
        int index_current = row * width + col;
        int index_flip = (row + 1) * width - col - 1;

        // Load current and flip pixel once
        RGB_8 pixel_current = img[index_current];
        RGB_8 pixel_flip = img[index_flip];

        // Compute gray value for current pixel
        float gray_value_current = 0.21f * pixel_current.r + 0.72f * pixel_current.g + 0.07f * pixel_current.b;
        pixel_current.r = gray_value_current;
        pixel_current.g = gray_value_current;
        pixel_current.b = gray_value_current;

        // Compute gray value for flip pixel before assignment to avoid overwriting issues
        float gray_value_flip = 0.21f * pixel_flip.r + 0.72f * pixel_flip.g + 0.07f * pixel_flip.b;

        // Set flip pixel to grayed current pixel
        img[index_flip] = pixel_current;

        // Set current pixel to original flip pixel
        pixel_flip.r = gray_value_flip;
        pixel_flip.g = gray_value_flip;
        pixel_flip.b = gray_value_flip;
        img[index_current] = pixel_flip;
    }
}