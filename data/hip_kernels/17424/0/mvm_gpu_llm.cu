#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

const double TOLERANCE = 1.0e-10;

/*
cgsolver with CUDA support solves the linear equation A*x = b where A is of size m x n
*/

__global__ void mvm_gpu(double *A_cuda, double *X_cuda, double *Y_cuda, int *m_locals_cuda, int *A_all_pos_cuda, int n, int nthreads) {
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (t < nthreads) {
        int startIdx = A_all_pos_cuda[t];
        int endIdx = startIdx + m_locals_cuda[t];
        
        // Pre-calculate the stride outside the inner loop
        for (int i = startIdx; i < endIdx; ++i) {
            double sum = 0.0;  // Use a local variable to accumulate the result
            for (int j = 0; j < n; ++j) {
                sum += A_cuda[i * n + j] * X_cuda[j];  // Accumulate sum
            }
            Y_cuda[i] = sum;  // Store the result
        }
    }
}