#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "MotionVec_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *new_image_dev = NULL;
hipMalloc(&new_image_dev, XSIZE*YSIZE);
float *old_image_dev = NULL;
hipMalloc(&old_image_dev, XSIZE*YSIZE);
uchar4 *Image_dev = NULL;
hipMalloc(&Image_dev, XSIZE*YSIZE);
int w = XSIZE;
int h = YSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
MotionVec<<<gridBlock, threadBlock>>>(new_image_dev,old_image_dev,Image_dev,w,h);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
MotionVec<<<gridBlock, threadBlock>>>(new_image_dev,old_image_dev,Image_dev,w,h);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
MotionVec<<<gridBlock, threadBlock>>>(new_image_dev,old_image_dev,Image_dev,w,h);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}