#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MotionVec(float *new_image_dev, float *old_image_dev, uchar4 *Image_dev, int w, int h)
{
    // Calculate global thread index
    const int ix = blockDim.x * blockIdx.x + threadIdx.x;
    const int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure within bounds for safety
    if (ix >= w || iy >= h) return;
    
    // Compute the difference squared
    float diff = old_image_dev[w * iy + ix] - new_image_dev[w * iy + ix];
    diff *= diff;

    // Threshold for motion detection
    float threshold = 5000.0f;

    // Update image where motion exceeds threshold
    if (diff > threshold)
    {
        Image_dev[w * iy + ix] = make_uchar4(0, 0, 255, 0); // Simplified assignment
    }
}