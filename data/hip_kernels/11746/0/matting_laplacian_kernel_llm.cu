#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 1e-4

__global__ void matting_laplacian_kernel(float *input, float *grad, int h, int w, int *CSR_rowIdx, int *CSR_colIdx, float *CSR_val, int N) {
    int size = h * w;
    int _id = blockIdx.x * blockDim.x + threadIdx.x;

    if (_id >= size) return;  // Early exit if _id is out of bounds

    int x = _id % w, y = _id / w;
    int id = x * h + y;

    // Binary search optimization: replace with simple lookup since CSR_rowIdx is sorted
    int index_start = CSR_rowIdx[id] - 1;
    int index_end = ((id + 1 < size) ? CSR_rowIdx[id + 1] : N) - 1;

    for (int i = index_start; i < index_end; i++) {
        int _colIdx = CSR_colIdx[i] - 1;
        float val = CSR_val[i];
        
        int _x = _colIdx / h, _y = _colIdx % h;
        int colIdx = _y * w + _x;

        // Access and compute each component once
        float input_val = 2 * val;
        grad[_id]         += input_val * input[colIdx];
        grad[_id + size]  += input_val * input[colIdx + size];
        grad[_id + 2*size] += input_val * input[colIdx + 2*size];
    }
    
    return;
}