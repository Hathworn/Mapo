#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 1e-4

__global__ void reconstruction_best_kernel(float *input, float *filtered_affine_model, float *filtered_best_output, int h, int w)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;

    if (id < size) {
        // Load frequently used data once
        float input_id = input[id];
        float input_id_size = input[id + size];
        float input_id_2size = input[id + 2 * size];
        float *affine_model_id = &filtered_affine_model[id * 12];

        // Compute outputs using shared affine model data
        double out1 = input_id_2size * affine_model_id[0] +
                      input_id_size * affine_model_id[1] +
                      input_id * affine_model_id[2] +
                      affine_model_id[3];
        double out2 = input_id_2size * affine_model_id[4] +
                      input_id_size * affine_model_id[5] +
                      input_id * affine_model_id[6] +
                      affine_model_id[7];
        double out3 = input_id_2size * affine_model_id[8] +
                      input_id_size * affine_model_id[9] +
                      input_id * affine_model_id[10] +
                      affine_model_id[11];

        // Store output results
        filtered_best_output[id] = out1;
        filtered_best_output[id + size] = out2;
        filtered_best_output[id + 2 * size] = out3;
    }
    return;
}