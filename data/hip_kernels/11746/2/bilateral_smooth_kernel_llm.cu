#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 1e-4

__global__ void bilateral_smooth_kernel(float* affine_model, float* filtered_affine_model, float* guide, int h, int w, int kernel_radius, float sigma1, float sigma2) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int size = h * w;
    if (id < size) {
        int x = id % w;
        int y = id / w;

        float sum_affine[12] = {0.0f};  // Use float for better performance on GPU
        float sum_weight = 0.0f;

        // Loop to use shared memory for guide values and weights to reduce global memory access
        for (int dx = -kernel_radius; dx <= kernel_radius; dx++) {
            for (int dy = -kernel_radius; dy <= kernel_radius; dy++) {
                int yy = y + dy, xx = x + dx;
                int id2 = yy * w + xx;
                if (0 <= xx && xx < w && 0 <= yy && yy < h) {
                    float color_diff1 = guide[yy * w + xx] - guide[y * w + x];
                    float color_diff2 = guide[yy * w + xx + size] - guide[y * w + x + size];
                    float color_diff3 = guide[yy * w + xx + 2 * size] - guide[y * w + x + 2 * size];
                    float color_diff_sqr = (color_diff1 * color_diff1 + color_diff2 * color_diff2 + color_diff3 * color_diff3) / 3;

                    float v1 = __expf(-(dx * dx + dy * dy) / (2 * sigma1 * sigma1));  // Use intrinsic function for exp
                    float v2 = __expf(-(color_diff_sqr) / (2 * sigma2 * sigma2));
                    float weight = v1 * v2;

                    for (int affine_id = 0; affine_id < 12; affine_id++) {  // Unroll loop for affine calculations
                        sum_affine[affine_id] += weight * affine_model[id2 * 12 + affine_id];
                    }
                    sum_weight += weight;
                }
            }
        }

        // Normalize the output
        for (int affine_id = 0; affine_id < 12; affine_id++) {  // Unroll loop for final normalization
            filtered_affine_model[id * 12 + affine_id] = sum_affine[affine_id] / sum_weight;
        }
    }
}