#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 256
#define EPS 1e-4

__device__ bool InverseMat4x4(double m_in[4][4], double inv_out[4][4]) {
    double m[16], inv[16];
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            m[i * 4 + j] = m_in[i][j];
        }
    }

    inv[0] = m[5] * m[10] * m[15] - m[5] * m[11] * m[14] - m[9] * m[6] * m[15] + m[9] * m[7] * m[14] + m[13] * m[6] * m[11] - m[13] * m[7] * m[10];
    inv[4] = -m[4] * m[10] * m[15] + m[4] * m[11] * m[14] + m[8] * m[6] * m[15] - m[8] * m[7] * m[14] - m[12] * m[6] * m[11] + m[12] * m[7] * m[10];
    inv[8] = m[4] * m[9] * m[15] - m[4] * m[11] * m[13] - m[8] * m[5] * m[15] + m[8] * m[7] * m[13] + m[12] * m[5] * m[11] - m[12] * m[7] * m[9];
    inv[12] = -m[4] * m[9] * m[14] + m[4] * m[10] * m[13] + m[8] * m[5] * m[14] - m[8] * m[6] * m[13] - m[12] * m[5] * m[10] + m[12] * m[6] * m[9];
    inv[1] = -m[1] * m[10] * m[15] + m[1] * m[11] * m[14] + m[9] * m[2] * m[15] - m[9] * m[3] * m[14] - m[13] * m[2] * m[11] + m[13] * m[3] * m[10];
    inv[5] = m[0] * m[10] * m[15] - m[0] * m[11] * m[14] - m[8] * m[2] * m[15] + m[8] * m[3] * m[14] + m[12] * m[2] * m[11] - m[12] * m[3] * m[10];
    inv[9] = -m[0] * m[9] * m[15] + m[0] * m[11] * m[13] + m[8] * m[1] * m[15] - m[8] * m[3] * m[13] - m[12] * m[1] * m[11] + m[12] * m[3] * m[9];
    inv[13] = m[0] * m[9] * m[14] - m[0] * m[10] * m[13] - m[8] * m[1] * m[14] + m[8] * m[2] * m[13] + m[12] * m[1] * m[10] - m[12] * m[2] * m[9];
    inv[2] = m[1] * m[6] * m[15] - m[1] * m[7] * m[14] - m[5] * m[2] * m[15] + m[5] * m[3] * m[14] + m[13] * m[2] * m[7] - m[13] * m[3] * m[6];
    inv[6] = -m[0] * m[6] * m[15] + m[0] * m[7] * m[14] + m[4] * m[2] * m[15] - m[4] * m[3] * m[14] - m[12] * m[2] * m[7] + m[12] * m[3] * m[6];
    inv[10] = m[0] * m[5] * m[15] - m[0] * m[7] * m[13] - m[4] * m[1] * m[15] + m[4] * m[3] * m[13] + m[12] * m[1] * m[7] - m[12] * m[3] * m[5];
    inv[14] = -m[0] * m[5] * m[14] + m[0] * m[6] * m[13] + m[4] * m[1] * m[14] - m[4] * m[2] * m[13] - m[12] * m[1] * m[6] + m[12] * m[2] * m[5];
    inv[3] = -m[1] * m[6] * m[11] + m[1] * m[7] * m[10] + m[5] * m[2] * m[11] - m[5] * m[3] * m[10] - m[9] * m[2] * m[7] + m[9] * m[3] * m[6];
    inv[7] = m[0] * m[6] * m[11] - m[0] * m[7] * m[10] - m[4] * m[2] * m[11] + m[4] * m[3] * m[10] + m[8] * m[2] * m[7] - m[8] * m[3] * m[6];
    inv[11] = -m[0] * m[5] * m[11] + m[0] * m[7] * m[9] + m[4] * m[1] * m[11] - m[4] * m[3] * m[9] - m[8] * m[1] * m[7] + m[8] * m[3] * m[5];
    inv[15] = m[0] * m[5] * m[10] - m[0] * m[6] * m[9] - m[4] * m[1] * m[10] + m[4] * m[2] * m[9] + m[8] * m[1] * m[6] - m[8] * m[2] * m[5];

    double det = m[0] * inv[0] + m[1] * inv[4] + m[2] * inv[8] + m[3] * inv[12];

    if (abs(det) < 1e-9) {
        return false;
    }

    det = 1.0 / det;

    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            inv_out[i][j] = inv[i * 4 + j] * det;
        }
    }

    return true;
}

__global__ void best_local_affine_kernel(float *output, float *input, float *affine_model, int h, int w, float epsilon, int kernel_radius) {
    int size = h * w;
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < size) {
        int x = id % w, y = id / w;

        double Mt_M[4][4] = {}; // 4x4
        double invMt_M[4][4] = {};
        double Mt_S[3][4] = {}; // RGB -> 1x4
        double A[3][4] = {};

        // Initialize matrices with identity and zeros where applicable
        for (int i = 0; i < 3; i++) {
            Mt_M[i][i] = 1e-3;
        }

        for (int dy = -kernel_radius; dy <= kernel_radius; dy++) {
            for (int dx = -kernel_radius; dx <= kernel_radius; dx++) {
                int xx = x + dx, yy = y + dy;
                int id2 = yy * w + xx;

                if (0 <= xx && xx < w && 0 <= yy && yy < h) {
                    // Input indices
                    int idx0 = id2;
                    int idx1 = id2 + size;
                    int idx2 = id2 + 2 * size;

                    // Update matrix Mt_M
                    Mt_M[0][0] += input[idx2] * input[idx2];
                    Mt_M[0][1] += input[idx2] * input[idx1];
                    Mt_M[0][2] += input[idx2] * input[idx0];
                    Mt_M[0][3] += input[idx2];

                    Mt_M[1][0] += input[idx1] * input[idx2];
                    Mt_M[1][1] += input[idx1] * input[idx1];
                    Mt_M[1][2] += input[idx1] * input[idx0];
                    Mt_M[1][3] += input[idx1];

                    Mt_M[2][0] += input[idx0] * input[idx2];
                    Mt_M[2][1] += input[idx0] * input[idx1];
                    Mt_M[2][2] += input[idx0] * input[idx0];
                    Mt_M[2][3] += input[idx0];

                    Mt_M[3][0] += input[idx2];
                    Mt_M[3][1] += input[idx1];
                    Mt_M[3][2] += input[idx0];
                    Mt_M[3][3] += 1;

                    // Update matrix Mt_S
                    Mt_S[0][0] += input[idx2] * output[idx2];
                    Mt_S[0][1] += input[idx1] * output[idx2];
                    Mt_S[0][2] += input[idx0] * output[idx2];
                    Mt_S[0][3] += output[idx2];

                    Mt_S[1][0] += input[idx2] * output[idx1];
                    Mt_S[1][1] += input[idx1] * output[idx1];
                    Mt_S[1][2] += input[idx0] * output[idx1];
                    Mt_S[1][3] += output[idx1];

                    Mt_S[2][0] += input[idx2] * output[idx0];
                    Mt_S[2][1] += input[idx1] * output[idx0];
                    Mt_S[2][2] += input[idx0] * output[idx0];
                    Mt_S[2][3] += output[idx0];
                }
            }
        }

        bool success = InverseMat4x4(Mt_M, invMt_M);

        // Matrix multiplication invMt_M * Mt_S = A
        for (int i = 0; i < 3; i++) {
            for (int j = 0; j < 4; j++) {
                double sum = 0.0;
                for (int k = 0; k < 4; k++) {
                    sum += invMt_M[j][k] * Mt_S[i][k];
                }
                A[i][j] = sum;
            }
        }

        // Assign calculated affine model values
        for (int i = 0; i < 3; i++) {
            for (int j = 0; j < 4; j++) {
                int affine_id = i * 4 + j;
                affine_model[12 * id + affine_id] = A[i][j];
            }
        }
    }
    return;
}