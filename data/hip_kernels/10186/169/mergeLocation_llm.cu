#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index within bounds before doing any computation
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];

        // Utilize shared memory for intermediate storage to avoid repeated global memory access
        __shared__ float shared_x[256];  // Assuming blockDim.x = 256, adjust according to actual block size
        __shared__ float shared_y[256];
        
        // Perform calculations and store in shared memory
        shared_x[threadIdx.x] = loc.x * scale;
        shared_y[threadIdx.x] = loc.y * scale;
        
        // Sync threads to ensure shared memory is populated
        __syncthreads();
        
        // Write back to global memory from shared memory
        x[ptidx] = shared_x[threadIdx.x];
        y[ptidx] = shared_y[threadIdx.x];
    }
}