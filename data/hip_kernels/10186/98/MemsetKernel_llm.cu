#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global thread index
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Ensure thread operates within bounds
    if (i < h && j < w)
    {
        // Calculate position once and directly assign value
        const int pos = i * w + j;
        image[pos] = value;
    }
}