#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    // Compute global thread indices
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Boundary guard
    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate the mapping with offset for the flow
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];
    float weight;

    // Macro for pixel contribution calculation
    #define ADD_TO_DEST(tx, ty, dx_factor, dy_factor)                \
        if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0))) {     \
            weight = dx_factor * dy_factor;                          \
            atomicAdd(dst + ty * image_stride + tx, value * weight); \
            atomicAdd(normalization_factor + ty * image_stride + tx, weight); \
        }

    // Calculate contributions to each of the target pixels
    ADD_TO_DEST(tx, ty, dx, dy)              // Bottom right
    ADD_TO_DEST(tx - 1, ty, (1.0f - dx), dy) // Bottom left
    ADD_TO_DEST(tx - 1, ty - 1, (1.0f - dx), (1.0f - dy)) // Upper left
    ADD_TO_DEST(tx, ty - 1, dx, (1.0f - dy)) // Upper right

    #undef ADD_TO_DEST
}