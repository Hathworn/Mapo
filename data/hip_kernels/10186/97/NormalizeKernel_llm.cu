#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Preload dimensions to avoid redundant global memory access
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Use conditional logic to prevent out-of-bound access
    if (i < h && j < w) 
    {
        const int pos = i * s + j;

        // Use a ternary operator for concise inversion logic
        float scale = normalization_factor[pos];
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

        // Perform in-place scaling operation
        image[pos] *= invScale;
    }
}