#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Imag function remains unchanged

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Calculate unique global thread index
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Exit if thread index is out of bounds
    if (i >= h || j >= w) return;

    // Precompute offsets for flow and image
    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Load u and v vectors for current pixel
    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Calculate bottom-left corner's target pixel coordinates
    float cx = u_ * time_scale + float(j) + 1.0f;
    float cy = v_ * time_scale + float(i) + 1.0f;

    // Round coordinates to nearest integer
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Load source pixel value
    float value = src[image_row_offset + j];

    // Check bounds and perform atomic addition
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}