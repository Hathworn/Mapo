#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void strided_access(float* a, int s, int max) {
    // Calculate the global index with stride
    int base_idx = idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride_idx = base_idx * s;

    // Loop using stride to avoid boundary check each iteration
    for (; stride_idx < max; stride_idx += s) {
        a[stride_idx] = a[stride_idx] + 1;
    }
}