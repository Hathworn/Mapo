#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce_v2(float* in, float* out, int n) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BX = blockDim.x;
    int i = bx * BX + tx;

    __shared__ float S[THEAD_MAX];

    S[tx] = (i < n) ? in[i] : 0;
    __syncthreads();

    // Use loop unrolling for optimization
    for (int s = BX / 2; s > 32; s >>= 1) {
        if (tx < s) {
            S[tx] += S[tx + s];
        }
        __syncthreads();
    }

    // Unrolling last warp
    if (tx < 32) {
        volatile float* vS = S;
        vS[tx] += vS[tx + 32];
        vS[tx] += vS[tx + 16];
        vS[tx] += vS[tx + 8];
        vS[tx] += vS[tx + 4];
        vS[tx] += vS[tx + 2];
        vS[tx] += vS[tx + 1];
    }

    if (tx == 0) {
        out[bx] = S[0];
    }
}