#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void prefixSumBackward(float* arr, int step) {
    // Improved indexing calculation
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int ii = i + 1;

    // Use of shared memory could be considered for further performance improvement
    if (ii <= n / step) {
        int idx = ii * step - 1;
        int halfStepIdx = ii * step - step / 2 - 1;

        // Use register variables for temporary storage
        int temp = arr[idx];
        arr[idx] += arr[halfStepIdx];
        arr[halfStepIdx] = temp;
    }
}