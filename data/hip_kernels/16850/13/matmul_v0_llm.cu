#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmul_v0(float* a, float* b, float* c, int n) {
    // C(nxn) = A(nxn) * B(nxn);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= n || j >= n) return;

    float c_ij = 0;

    // Use shared memory to load A and B tiles
    __shared__ float Asub[TILE_SIZE][TILE_SIZE];
    __shared__ float Bsub[TILE_SIZE][TILE_SIZE];

    for (int tile = 0; tile < n / TILE_SIZE; ++tile) {
        // Load data into shared memory
        Asub[threadIdx.y][threadIdx.x] = a[n * j + tile * TILE_SIZE + threadIdx.x];
        Bsub[threadIdx.y][threadIdx.x] = b[n * (tile * TILE_SIZE + threadIdx.y) + i];

        __syncthreads(); // Wait for all threads to load their parts

        // Accumulate the results
        for (int k = 0; k < TILE_SIZE; ++k) {
            c_ij += Asub[threadIdx.y][k] * Bsub[k][threadIdx.x];
        }

        __syncthreads(); // Wait for all threads to finish calculation
    }

    c[n * j + i] = c_ij; // Store the result
}
```
