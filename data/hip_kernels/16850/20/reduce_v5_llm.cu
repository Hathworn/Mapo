#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__inline__ __device__ void warp_reduce(float* S, int tx) {
    // Perform warp reduction without __syncthreads() for faster execution
    S[tx] += S[tx + 32];
    S[tx] += S[tx + 16];
    S[tx] += S[tx + 8];
    S[tx] += S[tx + 4];
    S[tx] += S[tx + 2];
    S[tx] += S[tx + 1];
}

__global__ void reduce_v5(float* in, float* out, int n) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int i = bx * (BX * 2) + tx;

    __shared__ float S[BX];

    // Improved coalesced access for higher memory throughput
    if (i < n) S[tx] = in[i] + ((i + BX) < n ? in[i + BX] : 0.0f);
    else S[tx] = 0.0f;
    __syncthreads();

    if (BX >= 1024) { if (tx < 512) S[tx] += S[tx + 512]; __syncthreads(); }
    if (BX >= 512) { if (tx < 256) S[tx] += S[tx + 256]; __syncthreads(); }
    if (BX >= 256) { if (tx < 128) S[tx] += S[tx + 128]; __syncthreads(); }
    if (BX >= 128) { if (tx < 64) S[tx] += S[tx + 64]; __syncthreads(); }

    if (tx < WARP_SIZE) warp_reduce(S, tx);

    if (tx == 0) out[bx] = S[0];
}