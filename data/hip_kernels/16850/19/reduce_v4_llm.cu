#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void warp_reduce(float* S, int tx) {
    S[tx] += S[tx + 32];
    S[tx] += S[tx + 16];
    S[tx] += S[tx + 8];
    S[tx] += S[tx + 4];
    S[tx] += S[tx + 2];
    S[tx] += S[tx + 1];
}

__global__ void reduce_v4(float* in, float* out, int n) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BX = blockDim.x; // same as THREAD_MAX
    int i = bx * (BX * 2) + tx;

    __shared__ float S[THREAD_MAX];

    // Improved memory coalescing and reduced thread divergence
    S[tx] = in[i] + (i + BX < n ? in[i + BX] : 0.0f);
    __syncthreads();

    // Efficient shared memory reduction
    for (int s = BX / 2; s > WARP_SIZE; s >>= 1) {
        if (tx < s) {
            S[tx] += S[tx + s];
        }
        __syncthreads();
    }

    // Warp-level reduction
    if (tx < WARP_SIZE) {
        warp_reduce(S, tx);
    }

    // Output the result for this block
    if (tx == 0) {
        out[bx] = S[0];
    }
}