#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transpose_v4(float* a, float* b, int n) {
    // Improved blockIdx calculation for better warp utilization
    int blockIdx_x = blockIdx.y;
    int blockIdx_y = (blockIdx.x + blockIdx.y) % gridDim.x;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int bx = blockIdx_x;
    int by = blockIdx_y;

    int i = bx * BX + tx;
    int j = by * BY + ty;

    // Use shared memory to reduce global memory access latency
    __shared__ float tile[BY][BX + 1]; // Minor modification to prevent bank conflict

    // Boundary check
    if (i < n && j < n) {
        tile[ty][tx] = a[j * n + i];
    }

    __syncthreads();

    // Transpose and write back with adjusted indices
    i = by * BY + tx;
    j = bx * BX + ty;

    // Boundary check for writing
    if (i < n && j < n) {
        b[j * n + i] = tile[tx][ty];
    }
}