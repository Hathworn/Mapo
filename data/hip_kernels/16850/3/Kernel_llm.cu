#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Kernel(int* a, int* b, int* c, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory effectively to avoid bank conflicts and redundant writes
    __shared__ int shared_mem[BLOCK_SIZE];
    if (i >= n) return;
    
    // Load data into a register and shared memory
    int reg = a[i] + b[i];
    shared_mem[threadIdx.x] = reg;
    __syncthreads();

    // Write from shared memory to global memory
    c[i] = shared_mem[threadIdx.x];
}