#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define TILE_SIZE 16

__global__ void matmul_v1(float* a, float* b, float* c, int n) {
    // C(nxn) = A(nxn) * B(nxn);

    __shared__ float A[TILE_SIZE][TILE_SIZE + 1];
    __shared__ float B[TILE_SIZE][TILE_SIZE + 1];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int i = bx * TILE_SIZE + tx;
    int j = by * TILE_SIZE + ty;

    // Initialize shared memory in a single assignment
    A[ty][tx] = 0.0f;
    B[ty][tx] = 0.0f;
    if (i >= n || j >= n) return;

    float c_ij = 0.0f;
    // Use ceil for loop iteration count
    int num_tiles = ceilf(float(n) / TILE_SIZE);

    for (int m = 0; m < num_tiles; m++) {
        // Load A and B into shared memory using stride access to ensure coalescing
        if (m * TILE_SIZE + tx < n && j < n) {
            A[ty][tx] = a[j * n + m * TILE_SIZE + tx];
        }
        if (m * TILE_SIZE + ty < n && i < n) {
            B[ty][tx] = b[(m * TILE_SIZE + ty) * n + i];
        }

        __syncthreads();

        // Unroll loop for better performance
        #pragma unroll
        for (int k = 0; k < TILE_SIZE; k++) {
            c_ij += A[ty][k] * B[k][tx];
        }
        __syncthreads();
    }

    // Check bounds before writing the result to global memory
    if (i < n && j < n) {
        c[n * j + i] = c_ij;
    }
}