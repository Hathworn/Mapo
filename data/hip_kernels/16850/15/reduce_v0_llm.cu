#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_v0(float* in, float* out, int n) {
    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int BX = blockDim.x; 
    int i = bx * BX + tx;

    __shared__ float S[THEAD_MAX];

    // Initialize shared memory with input data if within bounds
    S[tx] = (i < n) ? in[i] : 0;
    __syncthreads();

    // Perform reduction in shared memory
    for (int s = BX / 2; s > 0; s >>= 1) { // Use stride halving
        if (tx < s) {
            S[tx] += S[tx + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tx == 0) {
        out[bx] = S[0];
    }
}