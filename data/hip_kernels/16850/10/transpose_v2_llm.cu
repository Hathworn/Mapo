#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_v2(float* a, float* b, int n) {
    // Calculate the column and row index for the tile
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    
    int col = bx * blockDim.x + tx;
    int row = by * blockDim.y + ty;

    // Declare shared memory tile
    __shared__ float tile[BY][BX];

    // Check boundary
    if (col < n && row < n) {
        // Load into shared memory
        tile[ty][tx] = a[row * n + col];
    }
    __syncthreads();

    // Transpose the block and calculate transposed index
    col = by * blockDim.y + tx;
    row = bx * blockDim.x + ty;

    // Write transposed result to output matrix
    if (col < n && row < n) {
        b[row * n + col] = tile[tx][ty];
    }
}