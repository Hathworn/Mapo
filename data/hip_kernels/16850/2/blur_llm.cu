#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blur(int* B,int* G,int* R, int* RB,int* RG,int* RR, int* K, int rows, int cols, int krows, int kcols) {

    int index = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x for flexibility

    if (index >= rows*cols) // Correct boundary condition
        return;

    int pixel_row = index / cols;
    int pixel_col = index % cols; // Use modulo for clarity

    int k_sum = 0;
    int k_center_row = (krows - 1) / 2;
    int k_center_col = (kcols - 1) / 2;

    int sumB = 0, sumG = 0, sumR = 0; // Accumulators for RGB values

    for (int i = 0; i < krows; i++) {
        for (int j = 0; j < kcols; j++) {
            int kr = i - k_center_row;
            int kc = j - k_center_col;

            int pr = pixel_row + kr;
            int pc = pixel_col + kc;

            if (pr >= 0 && pr < rows && pc >= 0 && pc < cols) {
                int idx = pr * cols + pc;
                int kval = K[i * kcols + j]; // Use i and j for accessing kernel

                k_sum += kval;
                sumB += B[idx] * kval;
                sumG += G[idx] * kval;
                sumR += R[idx] * kval;
            }
        }
    }

    if (k_sum != 0) { // Avoid division by zero
        RB[index] = sumB / k_sum; // Normalized pixel values
        RG[index] = sumG / k_sum;
        RR[index] = sumR / k_sum;
    }
}