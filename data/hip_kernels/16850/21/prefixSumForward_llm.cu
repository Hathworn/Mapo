#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prefixSumForward(float* arr, int step, int n) {
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int BX = blockDim.x;
    int i = bx * BX + tx;
    int ii = i + 1;

    // Ensure index is within bounds and has work to do
    if (ii > n || ii <= n / float(step)) return;

    // Optimize memory access pattern
    arr[ii * step - 1] += arr[ii * step - step / 2 - 1];

    if (step == n && n - 1 == ii * step - 1) {
        arr[ii * step] = arr[ii * step - 1];
        arr[ii * step - 1] = 0;
    }
}