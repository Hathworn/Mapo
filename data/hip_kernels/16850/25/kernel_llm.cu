#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(int* arr, int offset_min, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use early exit strategy to reduce divergence
    if (i >= n || i < 0) return;

    // Efficient array access
    int idx = i + offset_min;
    arr[idx] += 1;
}