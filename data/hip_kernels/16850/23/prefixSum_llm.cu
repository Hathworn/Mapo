#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void prefixSum(float* arr, int step) {

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread doesn't go out of bounds
    if (i < step) return;

    // Use shared memory to reduce global memory accesses and increase memory coalescing
    extern __shared__ float shared_arr[];

    // Load data into shared memory
    shared_arr[threadIdx.x] = arr[i - step];
    __syncthreads();

    // Compute prefix sum using shared memory
    arr[i] += shared_arr[threadIdx.x];
}