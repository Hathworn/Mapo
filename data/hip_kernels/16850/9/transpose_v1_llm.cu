#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_v1(float* a, float* b, int n) {
    // Use shared memory for efficient access and reduce global memory accesses
    __shared__ float tile[32][32];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Load elements into shared memory tile
    if(x < n && y < n) {
        tile[threadIdx.y][threadIdx.x] = a[y * n + x];
    }
    
    __syncthreads(); // Ensure all threads have written to shared memory

    // Transpose and write back to global memory from shared memory tile
    x = blockIdx.y * blockDim.y + threadIdx.x;
    y = blockIdx.x * blockDim.x + threadIdx.y;

    if(x < n && y < n) {
        b[x * n + y] = tile[threadIdx.x][threadIdx.y];
    }
}