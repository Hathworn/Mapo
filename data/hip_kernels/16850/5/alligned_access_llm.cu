#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void alligned_access(float* a, int max) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < max) {
        // Use single if-statement to avoid out of bounds calculation
        a[idx] += 1.0f;
    }
}