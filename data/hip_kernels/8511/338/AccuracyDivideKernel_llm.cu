#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel to use block and thread indices for better parallel execution
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        *accuracy /= N;  // Perform division only once using a single thread
    }
}