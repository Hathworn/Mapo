#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void init_render_buffer(int64_t* render_buffer, const uint32_t qw_count) {
    const uint32_t start = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t step = blockDim.x * gridDim.x;
    for (uint32_t i = start; i < qw_count; i += step) {
        render_buffer[i] = EMPTY_KEY_64;  // Initialize element to EMPTY_KEY_64
    }
}

__global__ void init_render_buffer_wrapper(int64_t* render_buffer, const uint32_t qw_count) {
    // Calculate thread's unique index across grid
    const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Efficient index stride across all threads
    for (uint32_t i = idx; i < qw_count; i += blockDim.x * gridDim.x) {
        render_buffer[i] = EMPTY_KEY_64;  // Assign EMPTY_KEY_64 to the buffer
    }
}