#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void columnarize_groups(int8_t* columnar_buffer, const int8_t* rowwise_buffer, const size_t row_count, const size_t col_count, const size_t* col_widths, const size_t row_size) {
    const auto thread_index =
        threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Ensure the thread index is within bounds
    if (thread_index >= row_count) return;

    auto read_ptr = rowwise_buffer + thread_index * row_size;
    auto col_base = columnar_buffer;

    // Loop through columns
    for (size_t i = 0; i < col_count; ++i) {
        auto col_offset = thread_index;
        
        // DRY principle applied for memory casting and copying
        switch (col_widths[i]) {
            case 8:
                *reinterpret_cast<int64_t*>(col_base + col_offset * 8) = 
                    *reinterpret_cast<const int64_t*>(read_ptr);
                break;
            case 4:
                *reinterpret_cast<int32_t*>(col_base + col_offset * 4) = 
                    *reinterpret_cast<const int32_t*>(read_ptr);
                break;
            default:
                // Safe default action, handle other sizes or do nothing; currently no action
                break;
        }
        col_base += col_widths[i] * row_count;
        read_ptr += col_widths[i]; // POSSIBLE ISSUE: No handling of padding
    }
}