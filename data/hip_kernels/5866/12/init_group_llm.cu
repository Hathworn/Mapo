#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_group(int8_t* groups, const size_t group_count, const size_t col_count, const size_t* col_widths, const size_t* init_vals) {
    const auto thread_index = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;
    if (thread_index >= group_count) {
        return;
    }

    size_t offset = 0;
    for (size_t i = 0; i < col_count; ++i) {
        if (thread_index < group_count) {  // Ensure valid access
            switch (col_widths[i]) {
                case 4:
                    *reinterpret_cast<uint32_t*>(groups + offset * group_count + thread_index * 4) =
                        *reinterpret_cast<const uint32_t*>(init_vals + i);
                    break;
                case 8:
                    reinterpret_cast<size_t*>(groups + offset * group_count)[thread_index] =
                        init_vals[i];
                    break;
                default:
                    break;
            }
        }
        offset += col_widths[i];
    }
}