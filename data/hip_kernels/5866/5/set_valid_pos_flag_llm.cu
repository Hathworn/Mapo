#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_valid_pos_flag(int32_t* pos_buff, const int32_t* count_buff, const int32_t entry_count) {
    // Cache the thread and grid information
    const int32_t start = threadIdx.x + blockDim.x * blockIdx.x;
    const int32_t step = blockDim.x * gridDim.x;
    
    // Cache data in registers for faster access
    int32_t tid = start;
    
    while (tid < entry_count) {
        if (count_buff[tid]) {
            pos_buff[tid] = VALID_POS_FLAG;
        }
        tid += step;  // Increment by the step size
    }
}