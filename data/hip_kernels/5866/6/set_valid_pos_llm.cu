#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_valid_pos(int32_t* pos_buff, int32_t* count_buff, const int32_t entry_count) {
    const int32_t start = threadIdx.x + blockDim.x * blockIdx.x;
    const int32_t step = blockDim.x * gridDim.x;

    // Unrolling the loop for increased performance
    for (int32_t i = start; i < entry_count; i += step * 4) {
        if (VALID_POS_FLAG == pos_buff[i]) {
            pos_buff[i] = !i ? 0 : count_buff[i - 1];
        }
        if (i + step < entry_count && VALID_POS_FLAG == pos_buff[i + step]) {
            pos_buff[i + step] = (i + step == 0) ? 0 : count_buff[i + step - 1];
        }
        if (i + 2 * step < entry_count && VALID_POS_FLAG == pos_buff[i + 2 * step]) {
            pos_buff[i + 2 * step] = (i + 2 * step == 0) ? 0 : count_buff[i + 2 * step - 1];
        }
        if (i + 3 * step < entry_count && VALID_POS_FLAG == pos_buff[i + 3 * step]) {
            pos_buff[i + 3 * step] = (i + 3 * step == 0) ? 0 : count_buff[i + 3 * step - 1];
        }
    }
}