#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/
extern "C"
__global__ void copy_u2l_dense(double* ret, int dim, int N) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (iy < dim && ix < iy && iy * dim + ix < N) { // Bounds check and axis swap condition
        int id_dest = iy * dim + ix;
        int id_src = ix * dim + iy;
        ret[id_dest] = ret[id_src];
    }
}
```
