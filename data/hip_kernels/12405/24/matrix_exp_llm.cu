#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void matrix_exp(double *A, double *C, unsigned int size) {
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Launch only one thread per matrix element
    if (index < size) {
        // Utilize built-in function for fast calculation
        C[index] = __exp10(A[index] / 2.302585092994046); // optimize exp with __exp10
    }
}