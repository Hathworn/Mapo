#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/
extern "C"
__global__ void matrix_floor(double *A, double *C, unsigned int size) {
    int index = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
    
    // Check boundary with index before accessing memory
    if (index < size) {
        // Directly assign the floor of A[index] to C[index]
        C[index] = floor(A[index]);
    }
}