#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**********************************
When updating a kernel or adding a new one,
please compile the ptx file and commit it:
nvcc -ptx -arch=sm_30 SystemML.cu
***********************************/

/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/

extern "C"
__global__ void matrix_sqrt(double *A, double *C, unsigned int size) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        C[index] = sqrt(A[index]);  // Use sqrt directly as it efficiently computes square root
    }
}