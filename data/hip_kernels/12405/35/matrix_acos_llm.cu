#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* Licensed to the Apache Software Foundation (ASF) under one
* or more contributor license agreements.  See the NOTICE file
* distributed with this work for additional information
* regarding copyright ownership.  The ASF licenses this file
* to you under the Apache License, Version 2.0 (the
* "License"); you may not use this file except in compliance
* with the License.  You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing,
* software distributed under the License is distributed on an
* "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
* KIND, either express or implied.  See the License for the
* specific language governing permissions and limitations
* under the License.
*/

/**********************************
When updating a kernel or adding a new one,
please compile the ptx file and commit it:
nvcc -ptx -arch=sm_30 SystemML.cu
***********************************/

/**
* Optimized kernel function to compute acos for each element
* @param A input array
* @param C output array
* @param size number of elements in the arrays
*/
extern "C"
__global__ void matrix_acos(double *A, double *C, unsigned int size) {
    // Block and thread indexing for parallelism
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride

    // Loop through data with stride to cover all elements
    for (int i = index; i < size; i += stride) {
        C[i] = acos(A[i]); // Compute acos for each element
    }
}