#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int power_modulo_fast(long a, long b, long m) {
    long result = 1;
    long x = a % m;

    for (long i = 1; i <= b; i <<= 1) {
        x %= m;
        if ((b & i) != 0) {
            result *= x;
            result %= m;
        }
        x *= x;
    }

    return result;
}

__device__ float generate(hiprandState* globalState, int ind) {
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform(&localState);
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void kernel(int p, bool* prime, hiprandState* globalState) {
    int i = blockIdx.x;
    int a = 0;

    // Utilize shared memory for faster access
    __shared__ bool shared_prime;
    if (threadIdx.x == 0) {
        shared_prime = *prime;
    }
    __syncthreads();

    while(shared_prime && i < PRECISION) {
        a = (generate(globalState, i % BLOCKS_NUBMER) * (p - 2)) + 1;
        if(power_modulo_fast(a, p - 1, p) == 1) {
            i += BLOCKS_NUBMER;
        } else {
            shared_prime = false;
        }
    }

    if (threadIdx.x == 0) {
        *prime = shared_prime;
    }
}