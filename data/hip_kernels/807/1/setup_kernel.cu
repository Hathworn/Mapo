#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
int id = blockIdx.x;
int sequence = id;
int offset = 0;
hiprand_init ( seed, sequence, offset, &state[id] );
}