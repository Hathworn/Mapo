#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_kernel (hiprandState *state, unsigned long seed)
{
    // Improved parallelization with thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < gridDim.x * blockDim.x) {
        hiprand_init(seed, id, 0, &state[id]);
    }
}