#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int GetVecIndex(int vecNumber, int dimCount, int *dimSizes, int measCount, int vecCount, int *dims)
{
    unsigned long int index = 0;

    for (int i = 0; i < dimCount; ++i)
        index += (unsigned long int)dimSizes[i] * (unsigned long int)dims[i * vecCount + vecNumber];

    return index;
}

__global__ void AddPackKernel(unsigned long int *codes, int *measures, int dimensionsCount, int *dimendionsSizes, int measuresCount, int currentCapacity, int fullCapacity, int packCount, int *packDimensions, int *packMeasures)
{
    int currentVec = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  // Precompute stride

    while (currentVec < packCount)
    {
        // Optimize redundant memory accesses
        int vecIndex = GetVecIndex(currentVec, dimensionsCount, dimendionsSizes, measuresCount, packCount, packDimensions);
        codes[currentCapacity + currentVec] = vecIndex;

        for (int i = 0; i < measuresCount; ++i)
        {
            int measureIndex = i * fullCapacity + currentCapacity + currentVec;
            measures[measureIndex] = packMeasures[i * packCount + currentVec];
        }

        currentVec += stride;
    }
}