#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KernelLBSSimple(int aCount, const int* b_global, int bCount, int* indices_global) {

    __shared__ int data_shared[NT * VT];

    int tid = threadIdx.x;
    int x[VT];

    // Load bCount elements from B into data_shared using coalesced access
    int chunk_size = (bCount + NT - 1) / NT;
    int start = tid * chunk_size;
    int end = min(start + chunk_size, bCount);
    for (int index = start; index < end; ++index) {
        data_shared[index] = b_global[index];
    }
    __syncthreads();

    // Each thread searches for its Merge Path partition.
    int diag = VT * tid;
    int begin = max(0, diag - bCount);
    int end = min(diag, aCount);
    while (begin < end) {
        int mid = (begin + end) >> 1;
        int aKey = mid;
        int bKey = data_shared[diag - 1 - mid];
        if (aKey < bKey) begin = mid + 1;
        else end = mid;
    }
    int mp = begin;

    // Sequentially search, comparing indices a to elements data_shared[b].
    int a = mp;
    int b = diag - a;

    #pragma unroll
    for (int i = 0; i < VT; ++i) {
        bool p;
        if (b >= bCount) p = true;
        else if (a >= aCount) p = false;
        else p = a < data_shared[b];

        if (p)
            data_shared[bCount + a++] = b - 1; // Advance A
        else
            ++b; // Advance B
    }
    __syncthreads();

    // Store all indices to global memory.
    for (int i = tid; i < aCount; i += NT)
        indices_global[i] = data_shared[bCount + i];
}