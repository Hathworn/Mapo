#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

#if defined (__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#ifndef glutCloseFunc
#define glutCloseFunc glutWMCloseFunc
#endif
#else
#endif

__global__ void SyncCells(int * __restrict__ i_cells, const int * __restrict__ o_cells, int size)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Avoid checking out of bounds every time by using a grid-stride loop
    for (; idx < size; idx += blockDim.x * gridDim.x) {
        i_cells[idx] = o_cells[idx];
    }
}