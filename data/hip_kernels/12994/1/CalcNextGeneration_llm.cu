#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef _WIN32
#  define WINDOWS_LEAN_AND_MEAN
#  define NOMINMAX
#  include <windows.h>
#endif

#if defined (__APPLE__) || defined(MACOSX)
#pragma clang diagnostic ignored "-Wdeprecated-declarations"
#ifndef glutCloseFunc
#define glutCloseFunc glutWMCloseFunc
#endif
#else
#endif

__device__ int CountAliveCells(int *i_cells, int idx, int width, int height)
{
    int alive = 0;
    int posY = idx / width;
    int posX = idx % width;

    for (int i = -1; i <= 1; i++)
    {
        for (int j = -1; j <= 1; j++)
        {
            int currPosX = (posX + i + width) % width; // Avoid negative values
            int currPosY = (posY + j + height) % height;

            int neigh = currPosY * width + currPosX;

            if (i == 0 && j == 0)
                continue;
            if (i_cells[neigh] == 1)
                alive++;
        }
    }
    return alive;
}

__global__ void CalcNextGeneration(int *i_cells, int *o_cells, int width, int height)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx >= width * height)
        return;

    int neighCount = CountAliveCells(i_cells, idx, width, height);

    // Optimize branching logic by combining into a single line
    o_cells[idx] = (neighCount == 3 || (neighCount == 2 && i_cells[idx] == 1));

    __syncthreads(); // Ensure all threads reach this point
}