#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateExCovX(double *e_x_cov_x, double gauss_d2, int valid_voxel_num)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Using a linear grid-stride loop for better optimization across threads and blocks
    for (; id < valid_voxel_num; id += blockDim.x * gridDim.x) {
        e_x_cov_x[id] *= gauss_d2;  // Perform operation on each valid voxel
    }
}