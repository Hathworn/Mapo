#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumScore(double *score, int full_size, int half_size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Calculate the number of iterations needed based on half_size
    int iterations = (half_size + stride - 1) / stride;

    for (int i = 0; i < iterations; ++i) {
        int idx = i * stride + index;
        if (idx < half_size) {
            score[idx] += (idx + half_size < full_size) ? score[idx + half_size] : 0;
        }
    }
}