#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeScoreList(int *starting_voxel_id, int *voxel_id, int valid_points_num, double *e_x_cov_x, double gauss_d1, double *score)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory for frequent access
    __shared__ double local_score[1024];

    for (int i = id; i < valid_points_num; i += stride) {
        double score_inc = 0;
        for (int vid = starting_voxel_id[i]; vid < starting_voxel_id[i + 1]; vid++) {
            double tmp_ex = e_x_cov_x[vid];
            // Check for valid values and accumulate score increment
            score_inc += (fabs(tmp_ex) <= 1.0 && tmp_ex == tmp_ex) ? -gauss_d1 * tmp_ex : 0;
        }
        local_score[threadIdx.x] = score_inc;

        // Copy local results to global memory
        __syncthreads();
        score[i] = local_score[threadIdx.x];
    }
}