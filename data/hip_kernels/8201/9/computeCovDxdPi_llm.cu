#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeCovDxdPi(int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *inverse_covariance, int voxel_num, double gauss_d1, double gauss_d2, double *point_gradients, double *cov_dxd_pi, int valid_voxel_num)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int row = blockIdx.y;
    int col = blockIdx.z;

    // Ensure threads are working within the meaningful indices for row and col
    if (row < 3 && col < 6) {
        // Reorganized pointers for better memory coalescing
        double *icov_row_base = inverse_covariance + row * 3 * voxel_num;
        double *cov_dxd_pi_tmp = cov_dxd_pi + (row * 6 + col) * valid_voxel_num;
        double *pg_base = point_gradients + col * valid_points_num;

        for (int i = id; i < valid_points_num; i += stride) {
            double pg0 = pg_base[i];
            double pg1 = pg_base[i + 6 * valid_points_num];
            double pg2 = pg_base[i + 12 * valid_points_num];

            // Combine the two loops for contiguous memory access
            int start_vid = starting_voxel_id[i];
            int end_vid = starting_voxel_id[i + 1];
            for (int j = start_vid; j < end_vid; j++) {
                int vid = voxel_id[j];
                cov_dxd_pi_tmp[j] = icov_row_base[vid] * pg0 + 
                                    icov_row_base[voxel_num + vid] * pg1 + 
                                    icov_row_base[2 * voxel_num + vid] * pg2;
            }
        }
    }
}