#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeScoreGradientList(float *trans_x, float *trans_y, float *trans_z, int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *centroid_x, double *centroid_y, double *centroid_z, int voxel_num, double *e_x_cov_x, double *cov_dxd_pi, double gauss_d1, int valid_voxel_num, double *score_gradients)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int col = blockIdx.y;
    
    if (col < 6) {
        double *sg = score_gradients + col * valid_points_num;
        double *cov_dxd_pi_mat0 = cov_dxd_pi + col * valid_voxel_num;
        double *cov_dxd_pi_mat1 = cov_dxd_pi_mat0 + 6 * valid_voxel_num;
        double *cov_dxd_pi_mat2 = cov_dxd_pi_mat1 + 6 * valid_voxel_num;
        
        // Process multiple points per thread to increase parallelism
        for (int i = id; i < valid_points_num; i += stride) {
#pragma unroll 4 // Unrolling to improve throughput
            for (int p = 0; p < 4 && i + p < valid_points_num; p++) {
                int pid = valid_points[i + p];
                double d_x = static_cast<double>(trans_x[pid]);
                double d_y = static_cast<double>(trans_y[pid]);
                double d_z = static_cast<double>(trans_z[pid]);
                double tmp_sg = 0.0;

                // Restrict calculations to ensure coalesced memory access
                for (int j = starting_voxel_id[i + p]; j < starting_voxel_id[i + p + 1]; j++) {
                    int vid = voxel_id[j];
                    double tmp_ex = e_x_cov_x[j];

                    // Optimize conditional check to avoid NaN evaluation
                    if (tmp_ex >= 0 && tmp_ex <= 1) {
                        tmp_ex *= gauss_d1;
                        // Precompute index calculation for performance
                        double dx = d_x - centroid_x[vid];
                        double dy = d_y - centroid_y[vid];
                        double dz = d_z - centroid_z[vid];

                        // Accumulate result
                        tmp_sg += (dx * cov_dxd_pi_mat0[j] + dy * cov_dxd_pi_mat1[j] + dz * cov_dxd_pi_mat2[j]) * tmp_ex;
                    }
                }

                sg[i + p] = tmp_sg; // Store result
            }
        }
    }
}