#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeExCovX(float *trans_x, float *trans_y, float *trans_z, int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *centr_x, double *centr_y, double *centr_z, double gauss_d1, double gauss_d2, double *e_x_cov_x, double *icov00, double *icov01, double *icov02, double *icov10, double *icov11, double *icov12, double *icov20, double *icov21, double *icov22)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Optimize loop by using shared memory
    __shared__ double s_centr_x[1024];
    __shared__ double s_centr_y[1024];
    __shared__ double s_centr_z[1024];
    __shared__ double s_icov00[1024];
    __shared__ double s_icov01[1024];
    __shared__ double s_icov02[1024];
    __shared__ double s_icov10[1024];
    __shared__ double s_icov11[1024];
    __shared__ double s_icov12[1024];
    __shared__ double s_icov20[1024];
    __shared__ double s_icov21[1024];
    __shared__ double s_icov22[1024];
    
    for (int i = id; i < valid_points_num; i += stride) {
        int pid = valid_points[i];
        double d_x = static_cast<double>(trans_x[pid]);
        double d_y = static_cast<double>(trans_y[pid]);
        double d_z = static_cast<double>(trans_z[pid]);

        for (int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
            int vid = voxel_id[j];

            // Load data into shared memory
            if (threadIdx.x == 0) {
                s_centr_x[vid] = centr_x[vid];
                s_centr_y[vid] = centr_y[vid];
                s_centr_z[vid] = centr_z[vid];
                s_icov00[vid] = icov00[vid];
                s_icov01[vid] = icov01[vid];
                s_icov02[vid] = icov02[vid];
                s_icov10[vid] = icov10[vid];
                s_icov11[vid] = icov11[vid];
                s_icov12[vid] = icov12[vid];
                s_icov20[vid] = icov20[vid];
                s_icov21[vid] = icov21[vid];
                s_icov22[vid] = icov22[vid];
            }
            __syncthreads();

            double t_x = d_x - s_centr_x[vid];
            double t_y = d_y - s_centr_y[vid];
            double t_z = d_z - s_centr_z[vid];

            e_x_cov_x[j] = exp(-gauss_d2 * ((t_x * s_icov00[vid] + t_y * s_icov01[vid] + t_z * s_icov02[vid]) * t_x
                                           + ((t_x * s_icov10[vid] + t_y * s_icov11[vid] + t_z * s_icov12[vid]) * t_y)
                                           + ((t_x * s_icov20[vid] + t_y * s_icov21[vid] + t_z * s_icov22[vid]) * t_z)) / 2.0);
            __syncthreads();
        }
    }
}