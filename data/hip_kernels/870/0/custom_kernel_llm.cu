#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Calculate d[n] = a[n]*b[n] + c[n]

__global__ void custom_kernel(float *a, float *b, float *c, float *d, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Loop unrolling to improve performance
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < N; i += stride) {
        d[i] = a[i] * b[i] + c[i];
    }
}