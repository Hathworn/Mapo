#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot(float *a, float *b, float *c)
{
    // Declare cache for intermediate results
    __shared__ float cache[threadsPerBlock];
    int cacheIndex = threadIdx.x;

    // Calculate global thread ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Initialize temporary sum
    float temp = 0.0;

    // Unroll loop for better performance
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }

    // Store sum in cache
    cache[cacheIndex] = temp;
    __syncthreads();

    // Perform block-level reduction using a simpler binary reduction algorithm
    for (int i = blockDim.x/2; i > 0; i /= 2) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    // Write final block result to global output array
    if (threadIdx.x == 0) {
        c[blockIdx.x] = cache[0];
    }
}