#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel for better parallelization
__global__ void addition(int *a, int *b, int *c, int N)
{
    // Calculate global thread index for efficient parallel computation
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds before performing the addition
    if (tid < N) {
        c[tid] = a[tid] + b[tid];
    }
}