#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void addition(int *a, int *b, int *c)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global index
    c[idx] = a[idx] + b[idx]; // Perform element-wise addition
}