#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define T_PER_BLOCK 16
#define MINF __int_as_float(0xff800000)

__global__ void resampleFloat4_Kernel(float4* d_output, unsigned int outputWidth, unsigned int outputHeight, const float4* d_input, unsigned int inputWidth, unsigned int inputHeight)
{
    // Calculate unique thread indices
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread is within output bounds
    if (x < outputWidth && y < outputHeight)
    {
        // Pre-compute scaling factors
        const float scaleWidth = __fdividef(inputWidth - 1, outputWidth - 1);
        const float scaleHeight = __fdividef(inputHeight - 1, outputHeight - 1);

        // Calculate input coordinates with rounding
        const unsigned int xInput = __float2uint_rd(x * scaleWidth);
        const unsigned int yInput = __float2uint_rd(y * scaleHeight);

        // Perform bounds check and assign output
        if (xInput < inputWidth && yInput < inputHeight) {
            d_output[y * outputWidth + x] = d_input[yInput * inputWidth + xInput];
        }
    }
}