#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define T_PER_BLOCK 16
#define MINF __int_as_float(0xff800000)

__global__ void computeIntensityGradientMagnitude_Kernel(float* d_output, const float* d_input, unsigned int width, unsigned int height)
{
    const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        d_output[y*width + x] = MINF;

        // Simplified bounds check
        if (x > 0 && x < width - 1 && y > 0 && y < height - 1)
        {
            float pos00 = d_input[(y - 1)*width + (x - 1)];
            float pos01 = d_input[y*width + (x - 1)];
            float pos02 = d_input[(y + 1)*width + (x - 1)];
            float pos10 = d_input[(y - 1)*width + x];
            float pos12 = d_input[(y + 1)*width + x];
            float pos20 = d_input[(y - 1)*width + (x + 1)];
            float pos21 = d_input[y*width + (x + 1)];
            float pos22 = d_input[(y + 1)*width + (x + 1)];

            // Early exit if any MINF is detected
            if (pos00 == MINF || pos01 == MINF || pos02 == MINF || pos10 == MINF ||
                pos12 == MINF || pos20 == MINF || pos21 == MINF || pos22 == MINF)
                return;

            // Calculate gradients
            float resU = pos20 - pos00 + 2.0f * (pos21 - pos01) + pos22 - pos02;
            float resV = pos02 - pos00 + 2.0f * (pos12 - pos10) + pos22 - pos20;

            // Write result
            d_output[y*width + x] = sqrt(resU * resU + resV * resV);
        }
    }
}