#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define T_PER_BLOCK 16
#define MINF __int_as_float(0xff800000)

__global__ void computeIntensityDerivatives_Kernel(float2* d_output, const float* d_input, unsigned int width, unsigned int height)
{
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        d_output[y * width + x] = make_float2(MINF, MINF);

        // Check boundaries to avoid shared memory usage
        if (x > 0 && x < width - 1 && y > 0 && y < height - 1)
        {
            float pos00 = d_input[(y - 1) * width + (x - 1)];
            float pos01 = d_input[y * width + (x - 1)];
            float pos02 = d_input[(y + 1) * width + (x - 1)];
            float pos10 = d_input[(y - 1) * width + x];
            float pos12 = d_input[(y + 1) * width + x];
            float pos20 = d_input[(y - 1) * width + (x + 1)];
            float pos21 = d_input[y * width + (x + 1)];
            float pos22 = d_input[(y + 1) * width + (x + 1)];

            // Early exit if any input is MINF
            if (pos00 != MINF && pos01 != MINF && pos02 != MINF &&
                pos10 != MINF && pos12 != MINF &&
                pos20 != MINF && pos21 != MINF && pos22 != MINF) 
            {
                float resU = (-1.0f) * pos00 + (1.0f) * pos20 +
                             (-2.0f) * pos01 + (2.0f) * pos21 +
                             (-1.0f) * pos02 + (1.0f) * pos22;
                resU /= 8.0f;

                float resV = (-1.0f) * pos00 + (-2.0f) * pos10 + (-1.0f) * pos20 +
                             (1.0f) * pos02 + (2.0f) * pos12 + (1.0f) * pos22;
                resV /= 8.0f;

                d_output[y * width + x] = make_float2(resU, resV);
            }
        }
    }
}