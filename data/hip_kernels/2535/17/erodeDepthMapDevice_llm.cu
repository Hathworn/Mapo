#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define T_PER_BLOCK 16
#define MINF __int_as_float(0xff800000)

__global__ void erodeDepthMapDevice(float* d_output, float* d_input, int structureSize, int width, int height, float dThresh, float fracReq)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= 0 && x < width && y >= 0 && y < height)
    {
        unsigned int count = 0;
        float oldDepth = d_input[y * width + x];

        // Unrolling and optimization for the kernel loop
        for (int i = -structureSize; i <= structureSize; i++)
        {
            for (int j = -structureSize; j <= structureSize; j++)
            {
                int nx = x + j; // Precompute x + j
                int ny = y + i; // Precompute y + i
                
                if (nx >= 0 && nx < width && ny >= 0 && ny < height) // Check bounds
                {
                    float depth = d_input[ny * width + nx];
                    if (depth == MINF || depth == 0.0f || fabs(depth - oldDepth) > dThresh)
                    {
                        count++;
                    }
                }
            }
        }

        unsigned int sum = (2 * structureSize + 1) * (2 * structureSize + 1);
        if ((float)count / (float)sum >= fracReq) {
            d_output[y * width + x] = MINF;
        }
        else {
            d_output[y * width + x] = oldDepth; // Use pre-read oldDepth
        }
    }
}