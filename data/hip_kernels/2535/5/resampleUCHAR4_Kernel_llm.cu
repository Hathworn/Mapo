#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define T_PER_BLOCK 16
#define MINF __int_as_float(0xff800000)

__global__ void resampleUCHAR4_Kernel(uchar4* d_output, unsigned int outputWidth, unsigned int outputHeight, const uchar4* d_input, unsigned int inputWidth, unsigned int inputHeight)
{
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Use shared memory for frequently used parameters
    __shared__ float scaleWidth;
    __shared__ float scaleHeight;

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        scaleWidth = (float)(inputWidth - 1) / (float)(outputWidth - 1);
        scaleHeight = (float)(inputHeight - 1) / (float)(outputHeight - 1);
    }
    __syncthreads();

    if (x < outputWidth && y < outputHeight)
    {
        const unsigned int xInput = (unsigned int)(x * scaleWidth + 0.5f);
        const unsigned int yInput = (unsigned int)(y * scaleHeight + 0.5f);

        if (xInput < inputWidth && yInput < inputHeight) {
            d_output[y * outputWidth + x] = d_input[yInput * inputWidth + xInput];
        }
    }
}