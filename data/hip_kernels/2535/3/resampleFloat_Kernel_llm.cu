#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define T_PER_BLOCK 16
#define MINF __int_as_float(0xff800000)

__global__ void resampleFloat_Kernel(float* d_output, unsigned int outputWidth, unsigned int outputHeight, const float* d_input, unsigned int inputWidth, unsigned int inputHeight) 
{
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Pre-calculate scales outside if to avoid redundant calculation
    const float scaleWidth = (float)(inputWidth - 1) / (float)(outputWidth - 1);
    const float scaleHeight = (float)(inputHeight - 1) / (float)(outputHeight - 1);

    if (x < outputWidth && y < outputHeight) 
    {
        // Use efficient intrinsic to reduce type conversion
        const unsigned int xInput = __float2uint_rn(x * scaleWidth);
        const unsigned int yInput = __float2uint_rn(y * scaleHeight);

        if (xInput < inputWidth && yInput < inputHeight) 
        {
            // Use shared memory or register to minimize global memory access bottleneck (optional)
            d_output[y * outputWidth + x] = d_input[yInput * inputWidth + xInput];
            // d_output[y*outputWidth + x] = bilinearInterpolationFloat(x*scaleWidth, y*scaleHeight, d_input, inputWidth, inputHeight);
        }
    }
}