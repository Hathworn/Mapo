#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_print(long const* p, int n)
{
    // Get the thread's global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Only print the data if the thread index is zero to avoid redundant prints
    if (idx == 0) {
        printf("long: %d ", n);
        for (int i = 0; i < n; i++) {
            printf("%ld ", *(p + i));
        }
    }
}