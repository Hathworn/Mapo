#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_print(int const* p, int n)
{
    // Use local thread index to optimize memory access
    int idx = threadIdx.x;
    if (idx < n) {
        printf("int: %d ", n);
        printf("%d ", p[idx]);
    }
}