#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Perform division operation safely
    if (N != 0) {
        *accuracy /= N; 
    }
}