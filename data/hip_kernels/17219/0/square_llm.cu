#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved global kernel function with parallelism
__global__ void square(int *array, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique global thread index
    if (tid < n)
        array[tid] *= array[tid]; // Perform square operation
}