#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory for better performance
    __shared__ float sharedAcc;

    // Only one thread needs to do the division
    if (threadIdx.x == 0) {
        sharedAcc = *accuracy / N;
    }
    __syncthreads();

    // Ensure the result is written back to global memory
    if (threadIdx.x == 0) {
        *accuracy = sharedAcc;
    }
}