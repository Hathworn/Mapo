#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    outi += m * n * batch_index;
    out += m * n * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Coalesced copy from dist to out and outi
    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            out[j*n+s] = dist[j*n+s];
            outi[j*n+s] = s;
        }
    }

    __syncthreads(); // Ensure all elements are copied before sorting

    // Perform selection sort for the first k elements
    for (int j = index; j < m; j += stride) {
        float *p_dist = out + j * n;
        for (int s = 0; s < k; ++s) {
            int min = s;
            float min_val = p_dist[s];
            // find the min more efficiently
            for (int t = s + 1; t < n; ++t) {
                if (p_dist[t] < min_val) {
                    min = t;
                    min_val = p_dist[t];
                }
            }
            // swap min-th and i-th element
            if (min != s) {
                float tmp = p_dist[min];
                p_dist[min] = p_dist[s];
                p_dist[s] = tmp;
                int tmpi = outi[j*n+min];
                outi[j*n+min] = outi[j*n+s];
                outi[j*n+s] = tmpi;
            }
        }
    }
}