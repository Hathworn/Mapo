#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NmDistanceGradKernel(int b, int n, const float * xyz1, int m, const float * xyz2, const float * grad_dist1, const int * idx1, float * grad_xyz1, float * grad_xyz2) {
    // Get block and thread indices
    int batch_idx = blockIdx.x;
    int thread_idx = threadIdx.x + blockIdx.y * blockDim.x;

    // Loop through batches
    for (int i = batch_idx; i < b; i += gridDim.x) {
        // Loop through points
        for (int j = thread_idx; j < n; j += blockDim.x * gridDim.y) {
            // Load xyz1 data
            float x1 = xyz1[(i * n + j) * 3 + 0];
            float y1 = xyz1[(i * n + j) * 3 + 1];
            float z1 = xyz1[(i * n + j) * 3 + 2];
            
            // Fetch corresponding index
            int j2 = idx1[i * n + j];

            // Load xyz2 data
            float x2 = xyz2[(i * m + j2) * 3 + 0];
            float y2 = xyz2[(i * m + j2) * 3 + 1];
            float z2 = xyz2[(i * m + j2) * 3 + 2];

            // Calculate gradient magnitude
            float g = grad_dist1[i * n + j] * 2;

            // Update gradients for xyz1 using atomic addition
            atomicAdd(&grad_xyz1[(i * n + j) * 3 + 0], g * (x1 - x2));
            atomicAdd(&grad_xyz1[(i * n + j) * 3 + 1], g * (y1 - y2));
            atomicAdd(&grad_xyz1[(i * n + j) * 3 + 2], g * (z1 - z2));

            // Update gradients for xyz2 using atomic addition
            atomicAdd(&grad_xyz2[(i * m + j2) * 3 + 0], -(g * (x1 - x2)));
            atomicAdd(&grad_xyz2[(i * m + j2) * 3 + 1], -(g * (y1 - y2)));
            atomicAdd(&grad_xyz2[(i * m + j2) * 3 + 2], -(g * (z1 - z2)));
        }
    }
}