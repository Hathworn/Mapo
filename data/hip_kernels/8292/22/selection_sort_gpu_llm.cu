#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized selection sort kernel
__global__ void selection_sort_gpu(int b, int n, int m, int k, float *dist, int *idx, float *val) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    idx += m * k * batch_index;
    val += m * k * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    __shared__ float shared_dist[1024];  // Assuming n is <= 1024, use shared memory for faster access

    float *p_dist;
    for (int j = index; j < m; j += stride) {
        p_dist = dist + j * n;
        
        for (int t = index; t < n; t += stride) {  // Copy data to shared memory
            shared_dist[t] = p_dist[t];
        }
        __syncthreads();
        
        // Selection sort for the first k elements using shared memory
        for (int s = 0; s < k; ++s) {
            int min = s;
            
            // Find the min, operate on shared memory
            for (int t = s + 1; t < n; ++t) {
                if (shared_dist[t] < shared_dist[min]) {
                    min = t;
                }
            }
            
            // Update idx and val
            idx[j * n + s] = min;
            val[j * n + s] = shared_dist[min];
            
            // Swap min-th and i-th element in shared memory
            float tmp = shared_dist[min];
            shared_dist[min] = shared_dist[s];
            shared_dist[s] = tmp;
        }
    }
}