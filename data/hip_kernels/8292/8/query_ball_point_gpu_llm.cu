#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, const float* radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    int batch_idx = blockIdx.x;
    int point_idx = threadIdx.x;
    if(batch_idx < b && point_idx < m) { // Ensure within bounds
        const float* current_xyz1 = xyz1 + batch_idx * n * 3;
        const float* current_xyz2 = xyz2 + batch_idx * m * 3;
        int* current_idx = idx + batch_idx * m * nsample;
        float x2 = current_xyz2[point_idx * 3 + 0];
        float y2 = current_xyz2[point_idx * 3 + 1];
        float z2 = current_xyz2[point_idx * 3 + 2];
        int cnt = 0;
        
        for (int k = 0; k < n && cnt < nsample; ++k) {
            float x1 = current_xyz1[k * 3 + 0];
            float y1 = current_xyz1[k * 3 + 1];
            float z1 = current_xyz1[k * 3 + 2];
            float d_sq = (x2-x1)*(x2-x1) + (y2-y1)*(y2-y1) + (z2-z1)*(z2-z1);
            if (d_sq < radius[0] * radius[0]) {
                if (cnt == 0) { // Set default indices
                    for (int l = 0; l < nsample; ++l)
                        current_idx[point_idx * nsample + l] = k;
                }
                current_idx[point_idx * nsample + cnt] = k;
                cnt++;
            }
        }
    }
}