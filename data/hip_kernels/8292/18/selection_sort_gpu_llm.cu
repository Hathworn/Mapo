#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    outi += m * n * batch_index;
    out += m * n * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Improved memory coalescing
    for (int j = index; j < m; j += stride) {
        for (int s = 0; s < n; ++s) {
            out[j * n + s] = dist[j * n + s];
            outi[j * n + s] = s;
        }
    }

    float *p_dist;
    for (int j = index; j < m; j += stride) {
        p_dist = out + j * n;
        // Optimized loop tiling for better cache usage
        for (int s = 0; s < k; ++s) {
            int min = s;
            for (int t = s + 1; t < n; ++t) {
                if (p_dist[t] < p_dist[min]) {
                    min = t;
                }
            }
            if (min != s) {
                // Swap elements efficiently
                float tmp = p_dist[min];
                p_dist[min] = p_dist[s];
                p_dist[s] = tmp;
                int tmpi = outi[j * n + min];
                outi[j * n + min] = outi[j * n + s];
                outi[j * n + s] = tmpi;
            }
        }
    }
}