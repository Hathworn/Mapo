#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_index = blockIdx.x;
    int sample_index = blockIdx.y * blockDim.x + threadIdx.x;  // Flatten threadIdx for better grid usage
    
    if (batch_index < b && sample_index < m * nsample) {
        int j = sample_index / nsample;
        int k = sample_index % nsample;
        int ii = idx[j * nsample + k];
        
        for (int l = 0; l < c; ++l) {
            atomicAdd(&grad_points[ii * c + l], grad_out[j * nsample * c + k * c + l]);  // Use atomic add for concurrent writes
        }
    }
}