#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, const float *radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    int batch_index = blockIdx.x;
    xyz1 += n * 3 * batch_index;
    xyz2 += m * 3 * batch_index;
    idx += m * nsample * batch_index;

    int j = blockDim.x * blockIdx.y + threadIdx.x;
    if (j >= m) return; // Exit if thread is out of bounds for m
    
    float x2 = xyz2[j * 3 + 0];
    float y2 = xyz2[j * 3 + 1];
    float z2 = xyz2[j * 3 + 2];
    int cnt = 0;

    for (int k = 0; k < n; ++k) {
        if (cnt == nsample) break; // Only pick the FIRST nsample points in the ball
        float x1 = xyz1[k * 3 + 0];
        float y1 = xyz1[k * 3 + 1];
        float z1 = xyz1[k * 3 + 2];
        float dx = x2 - x1;
        float dy = y2 - y1;
        float dz = z2 - z1;
        float d = sqrtf(dx * dx + dy * dy + dz * dz);

        if (d < radius[0]) {
            if (cnt == 0) { // Initialize indices to k
                for (int l = 0; l < nsample; ++l) {
                    idx[j * nsample + l] = k;
                }
            }
            idx[j * nsample + cnt] = k;
            cnt += 1;
        }
    }
}