#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    // Calculate global index
    int batch_idx = blockIdx.x;
    int m_idx = blockIdx.y;
    int sample_idx = threadIdx.x;

    if (batch_idx < b && m_idx < m && sample_idx < nsample) {
        int ii = idx[m_idx * nsample + sample_idx];
        
        // Unroll the loop over 'c' where possible and use shared memory if necessary
        for (int l = 0; l < c; ++l) {
            out[m_idx * nsample * c + sample_idx * c + l] = points[ii * c + l];
        }
    }
}