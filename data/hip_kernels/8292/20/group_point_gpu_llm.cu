#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    int index = threadIdx.x;
    
    // Calculate offset for batch
    points += n * c * batch_index;
    idx += m * nsample * batch_index;
    out += m * nsample * c * batch_index;
    
    // Use parallel threads over samples in each iteration
    for (int j = 0; j < m; ++j) {
        for (int l = 0; l < c; ++l) {
            for (int k = index; k < nsample; k += blockDim.x) {
                int ii = idx[j * nsample + k];
                out[j * nsample * c + k * c + l] = points[ii * c + l];
            }
        }
    }
}