#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_index = blockIdx.x;
    int index = threadIdx.x + blockDim.x * blockIdx.y;
    
    if (index < m * nsample) {
        int j = index / nsample;
        int k = index % nsample;
        
        idx += m * nsample * batch_index;
        grad_out += m * nsample * c * batch_index;
        grad_points += n * c * batch_index;
        
        int ii = idx[j * nsample + k];
        for (int l = 0; l < c; ++l) {
            atomicAdd(&grad_points[ii * c + l], grad_out[j * nsample * c + k * c + l]); // Use atomicAdd to avoid race conditions
        }
    }
}