#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, const float *radius, int nsample, const float *xyz1, const float *xyz2, int *idx, int *pts_cnt) {
    int batch_index = blockIdx.x;
    xyz1 += n * 3 * batch_index;
    xyz2 += m * 3 * batch_index;
    idx += m * nsample * batch_index;
    pts_cnt += m * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;
    
    // Pre-compute radius square for comparison
    float r2 = radius[0] * radius[0];

    for (int j = index; j < m; j += stride) {
        float x2 = xyz2[j * 3 + 0];
        float y2 = xyz2[j * 3 + 1];
        float z2 = xyz2[j * 3 + 2];
        int cnt = 0;
        
        for (int k = 0; k < n && cnt < nsample; ++k) {
            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            
            // Calculate distance squared and compare with pre-computed radius squared
            float d2 = (x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1);
            
            if (d2 < r2) {
                if (cnt == 0) {
                    int nsample3 = nsample * j;
                    for (int l = 0; l < nsample; ++l)
                        idx[nsample3 + l] = k;
                }
                idx[j * nsample + cnt] = k;
                cnt += 1;
            }
        }
        
        pts_cnt[j] = cnt;
    }
}