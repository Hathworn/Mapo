#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, const float *radius, int nsample, const float *xyz1, const float *xyz2, int *idx) {
    int batchIdx = blockIdx.x; // Use blockIdx for batch processing
    int threadIdxInBlock = threadIdx.x;

    xyz1 += n * 3 * batchIdx; // Fix batch offset for input point set
    xyz2 += m * 3 * batchIdx; // Fix batch offset for query point set
    idx += m * nsample * batchIdx; // Fix batch offset for indices

    for (int j = threadIdxInBlock; j < m; j += blockDim.x) { // Use all threads for processing m query points in parallel
        int cnt = 0;
        float x2 = xyz2[j * 3 + 0];
        float y2 = xyz2[j * 3 + 1];
        float z2 = xyz2[j * 3 + 2];

        for (int k = 0; k < n && cnt < nsample; ++k) { // Combine loop condition to reduce checks
            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            float d = sqrtf((x2 - x1) * (x2 - x1) + (y2 - y1) * (y2 - y1) + (z2 - z1) * (z2 - z1));

            if (d < radius[0]) {
                if (cnt == 0) {
                    for (int l = 0; l < nsample; ++l)
                        idx[j * nsample + l] = k; // Fill with initial indices
                }
                idx[j * nsample + cnt] = k;
                cnt++;
            }
        }
    }
}