#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n * c * batch_index;
    idx += m * nsample * batch_index;
    out += m * nsample * c * batch_index;

    int index = threadIdx.x + blockIdx.y * blockDim.x; // Combine blockDim.y for more threads
    int stride = blockDim.x * gridDim.y; // Update stride for new configuration

    // Loop with new stride to utilize more threads
    for (int j = index; j < m * nsample; j += stride) {
        int sample_idx = j % nsample;
        int out_idx = j * c;
        int ii = idx[out_idx];
        for (int l = 0; l < c; ++l) {
            out[out_idx + l] = points[ii * c + l];
        }
    }
}