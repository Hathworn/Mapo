#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize daxbyAdd kernel function
__global__ void daxbyAdd(const float *A, const float *B, float *C, float x, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        // Use registers instead of global memory for intermediate calculations
        float a_val = A[i];
        float b_val = B[i];
        C[i] = a_val * x + b_val;
    }
}