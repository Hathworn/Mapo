#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use atomic operation to handle potential conflicts in multi-thread scenarios.
    atomicExch(accuracy, *accuracy / N);
}