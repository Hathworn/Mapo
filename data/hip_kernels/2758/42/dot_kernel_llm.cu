#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = gridDim.x * blockDim.x;
    for (; index < batch * n * size; index += totalThreads) { // Modularize parallel work
        int f1 = (index / size) / n;
        int f2 = (index / size) % n;
        if (f2 <= f1) continue;

        float sum = 0;
        float norm1 = 0;
        float norm2 = 0;

        for (int i = 0; i < size; ++i) {
            int i1 = f1 * size + i;
            int i2 = f2 * size + i;
            sum += output[i1] * output[i2];
            norm1 += output[i1] * output[i1];
            norm2 += output[i2] * output[i2];
        }

        norm1 = sqrt(norm1);
        norm2 = sqrt(norm2);
        float norm = norm1 * norm2;
        sum = sum / norm;

        for (int i = 0; i < size; ++i) {
            int i1 = f1 * size + i;
            int i2 = f2 * size + i;
            atomicAdd(&delta[i1], - scale * sum * output[i2] / norm); // Use atomic for concurrent writes
            atomicAdd(&delta[i2], - scale * sum * output[i1] / norm);
        }
    }
}