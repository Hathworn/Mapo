#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_sam_kernel(float *in_w_h_c_delta, int size, int channel_size, float *in_scales_c, float *out_from_delta, float *in_from_output, float *out_state_delta)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        // Use local variables to avoid repetitive global memory access
        float delta = in_w_h_c_delta[index];
        float from_output = in_from_output[index];
        float scale = in_scales_c[index];

        // Perform the operations with minimized global reads
        out_state_delta[index] += delta * from_output;
        out_from_delta[index] += scale * delta;
    }
}