#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within array bounds
    if (i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff); // Use built-in fabsf for fast absolute value calculation

        // Use ternary operator for better branch prediction
        error[i] = (abs_val < 1) ? (diff * diff) : (2 * abs_val - 1);
        delta[i] = (abs_val < 1) ? diff : ((diff < 0) ? -1 : 1);
    }
}