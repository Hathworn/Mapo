#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;

    extern __shared__ float shared_mean[];  // Use shared memory for mean

    float mean = 0.0f;
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        mean += fabs(weights[f*size + i]);
    }

    shared_mean[threadIdx.x] = mean;
    __syncthreads();

    // Reduce mean across block
    if (threadIdx.x == 0) {
        float total_mean = 0.0f;
        for (int i = 0; i < blockDim.x; ++i) {
            total_mean += shared_mean[i];
        }
        shared_mean[0] = total_mean / size;
    }
    __syncthreads();

    mean = shared_mean[0];

    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
    }
}