#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index more efficiently 
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Check if index is within bounds
    if (i < N) {
        X[i * INCX] = ALPHA;
    }
}