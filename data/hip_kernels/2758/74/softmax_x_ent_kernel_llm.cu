#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Streamlined single block index calculation

    if (i < n) {
        float t = truth[i];
        float p = pred[i];
        error[i] = (t) ? -logf(p) : 0; // Use logf for potentially faster computation in CUDA
        delta[i] = t - p;
    }
}