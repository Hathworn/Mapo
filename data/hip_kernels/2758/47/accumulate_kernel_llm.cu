#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Use shared memory for partial sums
    extern __shared__ float shared_sum[];
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Check bounds
    if (i >= groups) return;

    // Initialize shared memory
    shared_sum[tid] = 0;

    // Perform accumulation
    for (int k = 0; k < n; ++k) {
        shared_sum[tid] += x[k * groups + i];
    }

    // Write back to global memory
    sum[i] = shared_sum[tid];
}