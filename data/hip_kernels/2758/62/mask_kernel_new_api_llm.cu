#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel_new_api(int n, float *x, float mask_num, float *mask, float val)
{
    // Compute global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid-stride loop for better utilization
    for (; i < n; i += blockDim.x * gridDim.x) {
        if (mask[i] == mask_num) {
            x[i] = val;
        }
    }
}