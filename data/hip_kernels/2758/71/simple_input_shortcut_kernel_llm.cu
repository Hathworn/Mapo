#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_input_shortcut_kernel(float *in, int size, float *add, float *out)
{
    // Calculate global thread index with simplified 1D grid logic
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride to allow execution for remaining threads beyond initial range
    int stride = blockDim.x * gridDim.x;
  
    // Loop to cover all elements beyond initial range
    for (; id < size; id += stride) {
        out[id] = in[id] + add[id];  // Perform addition
    }
}