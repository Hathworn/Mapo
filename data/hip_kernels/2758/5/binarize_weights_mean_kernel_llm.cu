#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_mean_kernel(float *weights, int n, int size, float *binary, float *mean_arr_gpu)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= n * size) return;  // Exit if idx exceeds total number of weights

    int f = idx / size;
    float mean = mean_arr_gpu[f];
    
    // Use ternary operation for efficiency
    binary[idx] = (weights[idx] > 0) ? mean : -mean;
}