#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary) 
{
    // Calculate global index using single grid dimension
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundaries
    if (i < n) 
    {
        // Use conditional assignment for binarization
        binary[i] = (x[i] >= 0.0f) ? 1.0f : -1.0f;
    }
}