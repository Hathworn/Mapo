#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid-stride loop for efficient global memory access
    for (; i < n; i += blockDim.x * gridDim.x) {
        if (mask[i] == mask_num) {
            x[i] = mask_num;
        }
    }
}