#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_channels_kernel(float *in_w_h_c, int size, int channel_size, float *scales_c, float *out)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        // Load values into registers for efficient computation
        float input_val = in_w_h_c[index];
        float scale_val = scales_c[index / channel_size];
        // Perform scaling operation
        out[index] = input_val * scale_val;
    }
}