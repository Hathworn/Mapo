#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void assisted_activation_kernel(float alpha, float *output, float *gt_gpu, float *a_avg_gpu, int size, int channels, int batches)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for caching repeated calculations
    extern __shared__ float shared_data[];
    float *shared_gt_gpu = shared_data;
    float *shared_a_avg_gpu = shared_data + blockDim.x;

    if (i < size * batches) {  // Ensure valid index access
        int xy = i % size;
        int b = i / size;

        shared_gt_gpu[threadIdx.x] = gt_gpu[i];  // Load data into shared memory
        shared_a_avg_gpu[threadIdx.x] = a_avg_gpu[i];

        __syncthreads();  // Synchronize threads to ensure memory is loaded

        if (b < batches) {
            for (int c = 0; c < channels; ++c) {
                int idx = xy + size * (c + channels * b);
                output[idx] += alpha * shared_gt_gpu[threadIdx.x] * shared_a_avg_gpu[threadIdx.x];
            }
        }
    }
}