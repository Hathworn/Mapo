#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index
    if (id >= size) return;

    // Pre-compute reused terms to reduce repeated calculations
    int temp_size = id;
    int i = temp_size % minw;
    int offset_w = temp_size / minw;
    int j = offset_w % minh;
    int offset_h = offset_w / minh;
    int k = offset_h % minc;
    int b = offset_h / minc;

    int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b)); // Calculate output index
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b)); // Calculate add index
    out[out_index] += add[add_index];
}