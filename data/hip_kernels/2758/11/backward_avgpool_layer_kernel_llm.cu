#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n * c) return; // Fix index calculation for correct boundary check

    int k = id % c; // Compute channel index
    int b = id / c; // Compute batch index

    int out_index = k + c * b; // Compute output delta index
    float scale = out_delta[out_index] / (w * h); // Precompute scale factor
    
    // Use of a single loop variable
    for(int i = 0; i < w * h; ++i) {
        int in_index = i + w * h * (k + b * c); // Compute input delta index
        in_delta[in_index] += scale; // Apply scaled gradient
    }
}