#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation
    if (index < N) // Check bounds directly
    {
        int f = (index / spatial) % filters;
        float var_sqrt = rsqrtf(variance[f] + 1e-6f); // Use rsqrtf for better performance
        x[index] = (x[index] - mean[f]) * var_sqrt;
    }
}