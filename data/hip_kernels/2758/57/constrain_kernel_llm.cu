#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop for better utilization
    for (; i < N; i += gridDim.x * blockDim.x) {
        X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i * INCX]));
    }
}