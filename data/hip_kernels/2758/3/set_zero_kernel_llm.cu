#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use stride loop to ensure all threads in a block work efficiently
__global__ void set_zero_kernel(float *src, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride
    for (; i < size; i += stride) // Loop with a stride
    {
        src[i] = 0;
    }
}