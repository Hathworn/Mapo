#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_of_mults_kernel(float *a1, float *a2, float *b1, float *b2, size_t size, float *dst)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        // Prefetch data to reduce global memory access
        float a1_val = a1[index];
        float a2_val = a2[index];
        float b1_val = b1[index];
        float b2_val = b2[index];
        dst[index] = a1_val * a2_val + b1_val * b2_val;
    }
}