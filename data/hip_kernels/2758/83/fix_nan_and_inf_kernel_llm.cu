#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fix_nan_and_inf_kernel(float *input, size_t size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (index < size) {
        float val = input[index];
        
        // Directly assign to input to avoid redundant memory read
        if (isnan(val) || isinf(val))
            input[index] = 1.0f / (index + 1);  // Avoid division by zero
    }
}