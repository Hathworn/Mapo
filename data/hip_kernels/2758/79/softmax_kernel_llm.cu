#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(int n, float *input, float temp, float *output) {
    int i;
    float sum = 0;
    float largest = -INFINITY;
    
    // Find the largest input value in parallel
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        float val = input[i];
        atomicMax((int*)&largest, __float_as_int(val));
    }
    __syncthreads();

    // Compute exponentials and accumulate sum in parallel
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        float e = exp((input[i] - largest) / temp);
        atomicAdd(&sum, e);
        output[i] = e;
    }
    __syncthreads();

    // Normalize the output in parallel
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        output[i] /= sum;
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output) {
    int b = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (b >= batch) return;
    softmax_device(n, input + b * offset, temp, output + b * offset);
}