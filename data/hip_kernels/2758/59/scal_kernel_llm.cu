#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll the loop to increase performance
    int stride = blockDim.x * gridDim.x;
    for (; i < N; i += stride) {
        X[i * INCX] *= ALPHA;
    }
}