#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device_new_api(float *input, int n, float temp, int stride, float *output)
{
    float sum = 0.0f;
    float largest = -INFINITY;

    // Find the largest value for numerical stability
    for (int i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = fmaxf(val, largest);
    }

    // Compute the exponentials and the sum
    for (int i = 0; i < n; ++i) {
        float e = expf(input[i * stride] / temp - largest / temp);
        sum += e;
        output[i * stride] = e;
    }

    // Normalize the output
    for (int i = 0; i < n; ++i) {
        output[i * stride] /= sum;
    }
}

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    float sum = 0.0f;
    float largest = -INFINITY;

    // Find the largest value for numerical stability
    for (int i = 0; i < n; ++i) {
        float val = input[i];
        largest = fmaxf(val, largest);
    }

    // Compute the exponentials and the sum
    for (int i = 0; i < n; ++i) {
        float e = expf(input[i] / temp - largest / temp);
        sum += e;
        output[i] = e;
    }

    // Normalize the output
    for (int i = 0; i < n; ++i) {
        output[i] /= sum;
    }
}

__global__ void softmax_kernel_new_api(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (id >= batch * groups) return;
    
    int b = id / groups;
    int g = id % groups;

    // Use temporary pointers for input and output based on offsets for better readability
    float *temp_input = input + b * batch_offset + g * group_offset;
    float *temp_output = output + b * batch_offset + g * group_offset;

    softmax_device_new_api(temp_input, n, temp, stride, temp_output);
}