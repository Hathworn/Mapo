#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_copy_kernel(int size, float *src, float *dst)
{
    // Calculate index using a single expression for better readability
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Using shared memory to improve performance by reducing global memory access
    __shared__ float shared_mem[256];

    if (index < size)
    {
        // Load from global to shared memory
        shared_mem[threadIdx.x] = src[index];
        __syncthreads();

        // Store from shared to global memory
        dst[index] = shared_mem[threadIdx.x];
    }
}