#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc_avg_activation_kernel(float *src, float *dst, int size, int channels, int batches)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= size * batches) return; // Early exit for out-of-bounds

    int xy = i % size;
    int b = i / size;

    float sum = 0.0f;
    // Use shared memory if possible or adjust loop for efficient memory access
    for (int c = 0; c < channels; ++c) {
        sum += src[xy + size * (c + channels * b)];
    }
    dst[i] = sum / channels;
}