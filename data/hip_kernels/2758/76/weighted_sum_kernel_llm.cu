#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Combine blockIdx calculations for better thread indexing
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a loop to process multiple elements per thread if threads are less than n
    for (int idx = i; idx < n; idx += gridDim.x * blockDim.x) {
        c[idx] = s[idx] * a[idx] + (1.0f - s[idx]) * (b ? b[idx] : 0.0f);
    }
}