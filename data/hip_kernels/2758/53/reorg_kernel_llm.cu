#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate the global index for the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Calculate input coordinates
    int in_w = i % w;
    int in_h = (i / w) % h;
    int in_c = (i / (w * h)) % c;
    int b = (i / (w * h * c)) % batch;

    // Calculate output channel and index transformations
    int out_c = c / (stride * stride);
    int offset = in_c / out_c;
    int c2 = in_c % out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Conditional forward or backward operation
    if (forward)
        out[out_index] = x[i];
    else
        out[i] = x[out_index];
}