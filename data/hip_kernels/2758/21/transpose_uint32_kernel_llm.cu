#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_uint32_kernel(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate row and column indices
    int i = index % src_h;
    int j = index / src_h;

    // Check bounds and perform transposition
    if (j < src_w && i < src_h) 
    {
        // Optimize memory access by aligning indices correctly
        dst[j * dst_align / 32 + i] = src[i * src_align + j];
    }
}