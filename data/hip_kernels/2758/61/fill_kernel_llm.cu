#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Process each strided element
    for(int stride = i; stride < N; stride += blockDim.x * gridDim.x) {
        X[stride * INCX] = ALPHA;
    }
}