#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplify 1D grid calculation
    if (index >= N) return;

    int f = (index / spatial) % filters;
    float inv_sqrt_var = rsqrtf(variance[f] + .000001f); // Use faster rsqrtf for reciprocal square root
    float x_minus_mean = x[index] - mean[f];

    float var_delta_term = variance_delta[f] * 2.0f * x_minus_mean / (spatial * batch);
    float mean_delta_term = mean_delta[f] / (spatial * batch);

    delta[index] = delta[index] * inv_sqrt_var + var_delta_term + mean_delta_term;
}