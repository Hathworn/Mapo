#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Use tile-friendly access pattern to improve coalesced memory access
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(; i < N; i += stride) {
        Y[i*INCY] = powf(X[i*INCX], ALPHA);
    }
}