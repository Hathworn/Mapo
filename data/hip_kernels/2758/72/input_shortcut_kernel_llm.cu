#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void input_shortcut_kernel(float *in, int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D grid and block
    if (id >= size) return;

    // Pre-calculate reused values to reduce repetitive operations
    int sample_w2 = sample * w2;
    int stride_w1 = stride * w1;
    int minwh = minw * minh;
    
    // Calculate indices based on flat id
    int b = id / (minc * minwh);
    int k = (id / minwh) % minc;
    int j = (id / minw) % minh;
    int i = id % minw;

    int out_index = i * sample + sample_w2 * (j + h2 * (k + c2 * b));
    int add_index = i * stride + stride_w1 * (j + h1 * (k + c1 * b));
    out[out_index] = in[out_index] + add[add_index];
}