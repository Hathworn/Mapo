#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Compute global thread index for 1D grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i >= N) return; // Boundary check

    // Unroll index calculations
    int b = i / (layers * spatial);
    int remainder = i % (layers * spatial);
    int in_c = remainder / spatial;
    int in_s = remainder % spatial;

    // Calculate offset indices
    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;

    // Conditional assignment based on 'forward' flag
    if (forward) {
        out[i2] = x[i1];
    } else {
        out[i1] = x[i2];
    }
}