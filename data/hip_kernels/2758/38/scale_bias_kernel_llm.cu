#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    // Use shared memory for biases to minimize global memory access
    __shared__ float shared_biases[1024]; 

    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if (threadIdx.x < n && filter == 0) {
        shared_biases[threadIdx.x] = biases[threadIdx.x];
    }

    __syncthreads();  // Ensure all biases are loaded before use

    if(offset < size) {
        output[(batch*n+filter)*size + offset] *= shared_biases[filter];
    }
}