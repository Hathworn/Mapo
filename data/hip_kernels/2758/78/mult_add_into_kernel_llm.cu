#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop unrolling: process 4 elements per thread when possible
    int stride = gridDim.x * blockDim.x; // Total number of threads in grid
    for (; i < n; i += stride)
    {
        c[i] += a[i] * b[i];
    }
}