#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    int stride = blockDim.x * gridDim.x;              // Calculate stride for efficient memory access

    for (int i = idx; i < N; i += stride) {           // Loop with stride to handle larger N sizes efficiently
        if ((X[i * INCX] * X[i * INCX]) < (ALPHA * ALPHA)) {
            X[i * INCX] = 0;
        }
    }
}