#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Precompute scale outside loop
    float scale = 1.F/(batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i >= filters) return;

    float sum = 0.0f; // Use register for sum to reduce global memory access

    // Unroll loops for better efficiency
    for (int j = 0; j < batch; ++j) {
        int index_base = j * filters * spatial + i * spatial;
        for (int k = 0; k < spatial; ++k) {
            sum += x[index_base + k];
        }
    }
    
    mean[i] = sum * scale; // Store the results back to global memory
}