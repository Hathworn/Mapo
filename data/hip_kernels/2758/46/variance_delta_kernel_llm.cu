#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Load mean and variance values once, to reduce global memory accesses
    float mean_i = mean[i];
    float variance_i = variance[i];
    
    float var_delta = 0;
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            var_delta += delta[index] * (x[index] - mean_i);
        }
    }
    variance_delta[i] = var_delta * -.5f * powf(variance_i + .000001f, -1.5f);
}