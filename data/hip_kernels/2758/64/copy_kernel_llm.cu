#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified 1D grid calculation
    int stride = blockDim.x * gridDim.x; // Compute stride for coalesced access

    // Loop over data with stride to ensure all elements are processed
    for (int index = i; index < N; index += stride) {
        Y[index * INCY + OFFY] = X[index * INCX + OFFX];
    }
}