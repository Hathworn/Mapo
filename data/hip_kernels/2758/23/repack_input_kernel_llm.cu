#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void repack_input_kernel(float *input, float *re_packed_input, int w, int h, int c)
{
    // Calculate the global index for this thread in the kernel
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate items per channel assuming w * h
    const int items_per_channel = w * h;

    // Determine the sub-index of the kernel processing within a single channel block (32 items)
    int c_pack = index % 32;
    // Compute the current processing index across channels
    int chan_index = index / 32;
    // Calculate current channel
    int chan = (chan_index * 32) % c;
    // Calculate relative position/index within items_per_channel
    int i = (chan_index * 32) / c;

    // Proceed only if within boundary limits
    if (i < items_per_channel)
    {
        // Read source and write to destination in a coalesced memory access pattern
        float src = input[(chan + c_pack) * items_per_channel + i];
        re_packed_input[chan * items_per_channel + i * 32 + c_pack] = src;
    }
}