#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float mean = 0.0f; // Use a local variable for intermediate accumulation
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            mean += delta[index]; // Accumulate into the local variable
        }
    }
    mean_delta[i] = mean * (-1.F / sqrtf(variance[i] + .000001f)); // Compute the final result
}