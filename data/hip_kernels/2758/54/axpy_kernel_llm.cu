#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Use grid-stride loop to improve efficiency for large N
    for (int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x * gridDim.y) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}