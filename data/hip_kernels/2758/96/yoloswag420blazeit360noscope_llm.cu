#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    // Improved calculation of unique thread ID for better readability
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread ID is within bounds before performing operations
    if (id < size) {
        input[id] = (rand[id] < prob) ? 0 : input[id] * scale;
    }
}