#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolve_gpu_kernel(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n, int size, int pad) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int x = index % in_w;
    int index2 = index / in_w;
    int y = index2 % in_h;
    int fil = index2 / in_h;

    if (fil < n) {
        int const output_index = fil * in_w * in_h + y * in_w + x;
        float sum = 0.0f;

        for (int chan = 0; chan < in_c; ++chan) {
            int const weights_pre_index = fil * in_c * size * size + chan * size * size;
            int const input_pre_index = chan * in_w * in_h;

            // Loop unrolling for filter size
            for (int f_y = 0; f_y < size; ++f_y) {
                int input_y = y + f_y - pad;
                if (input_y < 0 || input_y >= in_h) continue;  // Skip invalid rows

                for (int f_x = 0; f_x < size; ++f_x) {
                    int input_x = x + f_x - pad;
                    if (input_x < 0 || input_x >= in_w) continue;  // Skip invalid cols

                    int input_index = input_pre_index + input_y * in_w + input_x;
                    int weights_index = weights_pre_index + f_y * size + f_x;

                    sum += input[input_index] * weights[weights_index];
                }
            }
        }
        output[output_index] = sum;
    }
}