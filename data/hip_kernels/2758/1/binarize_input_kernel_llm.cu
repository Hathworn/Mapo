#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = blockIdx.x * blockDim.x + threadIdx.x;
    if (s >= size) return;

    // Pre-compute mean for s
    float mean = 0;
    for(int i = 0; i < n; ++i) {
        mean += fabsf(input[i * size + s]);
    }
    mean /= n;

    // Binarize input based on mean
    for(int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}