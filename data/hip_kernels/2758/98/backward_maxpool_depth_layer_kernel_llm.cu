#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_depth_layer_kernel(int n, int w, int h, int c, int batch, float *delta, float *prev_delta, int *indexes)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation
    if (id < n) // Reverse if condition for early exit
    {
        int index = indexes[id];
        atomicAdd(&prev_delta[index], delta[id]); // Use atomic operation to avoid race conditions
    }
}