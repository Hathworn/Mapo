#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void is_nan_or_inf_kernel(float *input, size_t size, int *pinned_return)
{
    // Using shared memory for reduction
    extern __shared__ int smem[];
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int tid = threadIdx.x;

    // Initialize shared memory
    smem[tid] = 0;

    // Check for NaN or Inf and store result
    if (index < size) {
        float val = input[index];
        if (isnan(val) || isinf(val)) {
            smem[tid] = 1;
        }
    }

    // Reduce in shared memory
    __syncthreads();
    if (tid == 0) {
        for (int i = 1; i < blockDim.x; ++i) {
            smem[0] |= smem[i];
        }
        
        // Use atomic operation to update global result
        if (smem[0]) {
            atomicOr(pinned_return, 1);
        }
    }
}