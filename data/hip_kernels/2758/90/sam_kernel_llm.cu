#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sam_kernel(float *in_w_h_c, int size, int channel_size, float *scales_c, float *out)
{
    // Calculate global index
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x; // Calculate stride for loop

    // Loop with stride to cover all elements
    for (int i = index; i < size; i += stride) {
        out[i] = in_w_h_c[i] * scales_c[i];
    }
}