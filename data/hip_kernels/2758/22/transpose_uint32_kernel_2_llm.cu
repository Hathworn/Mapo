#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transpose_uint32_kernel_2(uint32_t *src, uint32_t *dst, int src_h, int src_w, int src_align, int dst_align)
{
    // Adjust shared memory size for improved performance alignment
    __shared__ uint32_t tmp[32 * 32]; 
    const int w_align = 32; 

    const int src_w_align = src_w + (32 - src_w % 32);

    // Calculate thread and block indices
    const int local_x = threadIdx.x % 32;
    const int local_y = threadIdx.x / 32;

    const int global_index = blockIdx.x;
    const int global_x_index = global_index % (src_w_align / 32);
    const int global_y_index = global_index / (src_w_align / 32);
    
    const int global_x = global_x_index * 32 + local_x;
    const int global_y = global_y_index * 32 + local_y;

    uint32_t val = 0;
    
    // Fetch data from global memory using conditional checks
    if (global_x < src_w && global_y < src_h) {
        val = src[global_y * src_align + global_x];
    }
    
    // Load data into shared memory with adjusted indices
    tmp[local_y * w_align + local_x] = val;
    __syncthreads();
    val = tmp[local_x * w_align + local_y];

    const int new_global_x = global_y_index * 32 + local_x;
    const int new_global_y = global_x_index * 32 + local_y;

    // Store result back into global memory
    if (new_global_x < src_h && new_global_y < src_w) {
        dst[new_global_y * (dst_align / 32) + new_global_x] = val;
    }
}