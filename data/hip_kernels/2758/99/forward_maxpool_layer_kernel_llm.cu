#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride_x, int stride_y, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + pad - size) / stride_y + 1;
    int w = (in_w + pad - size) / stride_x + 1;
    int c = in_c;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    int out_index = j + w * (i + h * (k + c * b));
    float max = -INFINITY;
    int max_i = -1;

    // Using temp variables for commonly calculated values to reduce access time
    int input_depth_step = in_h * in_w;
    int input_batch_step = in_c * input_depth_step;

    for (int l = 0; l < size; ++l) {
        for (int m = 0; m < size; ++m) {
            int cur_h = h_offset + i * stride_y + l;
            int cur_w = w_offset + j * stride_x + m;
            int index = cur_w + in_w * (cur_h + k * in_h + b * input_depth_step);
            int valid = (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w);
            float val = (valid != 0) ? input[index] : -INFINITY;

            // Using ternary operation to simplify the max comparison and update
            if (val > max) {
                max = val;
                max_i = index;
            }
        }
    }
    output[out_index] = max;
    indexes[out_index] = max_i;
}