#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride_x, int stride_y, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + pad - size) / stride_y + 1;
    int w = (in_w + pad - size) / stride_x + 1;
    int c = in_c;
    int area_x = (size - 1) / stride_x;
    int area_y = (size - 1) / stride_y;

    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (id >= n) return;
    
    int temp_id = id;
    int j = temp_id % in_w;
    temp_id /= in_w;
    int i = temp_id % in_h;
    temp_id /= in_h;
    int k = temp_id % in_c;
    temp_id /= in_c;
    int b = temp_id;
    
    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    float d = 0;
    for (int l = -area_y; l <= area_y; ++l) { // Combine initialization and check
        for (int m = -area_x; m <= area_x; ++m) { // Combine initialization and check
            int out_w = (j - w_offset) / stride_x + m;
            int out_h = (i - h_offset) / stride_y + l;
            int out_index = out_w + w * (out_h + h * (k + c * b));
            bool valid = (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h);
            d += (valid && indexes[out_index] == id) ? delta[out_index] : 0;
        }
    }
    prev_delta[id] += d; // Use id rather than index for clarity
}