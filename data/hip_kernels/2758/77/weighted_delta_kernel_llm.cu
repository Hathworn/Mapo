#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Flatten block index computation
    int gridStride = blockDim.x * gridDim.x;        // Calculate stride for grid-stride looping

    for (; i < n; i += gridStride) {                // Use grid-stride loop for better occupancy
        if (da) da[i] += dc[i] * s[i];
        db[i] += dc[i] * (1 - s[i]);
        ds[i] += dc[i] * (a[i] - b[i]);             // Simplify the calculation of ds[i]
    }
}