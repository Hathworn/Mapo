#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_add_kernel(int N, float ALPHA, float BETA, float *X, int INCX) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified block index calculation
    int stride = blockDim.x * gridDim.x;              // Calculate stride for skipping blocks
    for (int i = idx; i < N; i += stride) {           // Loop through all elements with a stride
        X[i * INCX] = X[i * INCX] * ALPHA + BETA;     // Efficient memory access pattern
    }
}