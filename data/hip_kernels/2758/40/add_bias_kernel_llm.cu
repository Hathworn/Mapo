#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    // Calculate unique index using flattened thread and block indices
    int index = blockIdx.z * gridDim.y * size + blockIdx.y * size + blockIdx.x * blockDim.x + threadIdx.x;

    // Use index to check and update the output
    if (index < n * size) {
        int filter = index / size % n;
        output[index] += biases[filter];
    }
}