#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global index more efficiently avoiding redundant operations
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for better performance with larger datasets
    for (; i < n; i += blockDim.x * gridDim.x) {
        float diff = truth[i] - pred[i];
        error[i] = diff * diff;
        delta[i] = diff;
    }
}