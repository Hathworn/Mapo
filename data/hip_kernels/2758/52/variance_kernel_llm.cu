#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Calculate scale factor outside the loop for efficiency
    float scale = 1.F/(batch * spatial - 1);
    // Determine global thread index
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (i >= filters) return;

    // Initialize shared variable for variance computation
    float var = 0.0f;

    // Loop through batch and spatial dimensions
    for(int j = 0; j < batch; ++j){
        for(int k = 0; k < spatial; ++k){
            int index = j * filters * spatial + i * spatial + k;
            float diff = x[index] - mean[i];
            var += diff * diff; // Avoid using powf for a slight performance gain
        }
    }
    
    // Scale the accumulated variance and store it
    variance[i] = var * scale;
}