#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    // Calculate global index
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Ensure index is within bounds
    if (idx >= N) return;

    // Compute index components
    int out_index = idx;
    int out_w = (out_index % (w * stride));
    out_index /= (w * stride);
    int out_h = (out_index % (h * stride));
    out_index /= (h * stride);
    int out_c = (out_index % c);
    out_index /= c;
    int b = (out_index % batch);

    // Compute input indices
    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_c = out_c;
    int in_index = b * w * h * c + in_c * w * h + in_h * w + in_w;

    // Apply forward or backward operation
    if (forward) {
        // Improve memory access by using shared memory or constant memory if possible
        out[idx] += scale * __ldg(&x[in_index]); // Use texture cache if applicable
    } else {
        atomicAdd(&x[in_index], scale * out[idx]);
    }
}