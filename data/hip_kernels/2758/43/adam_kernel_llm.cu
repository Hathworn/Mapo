#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify index calculation

    if (index < N)  // Reverse logic for efficiency
    {
        // Pre-compute powers outside the loop for performance
        float invB1T = 1.f / (1.f - powf(B1, t)); 
        float invB2T = 1.f / (1.f - powf(B2, t));
        
        float mhat = m[index] * invB1T;  // Use multiplication instead of division
        float vhat = v[index] * invB2T;  // Use multiplication instead of division

        x[index] += rate * mhat / (sqrtf(vhat) + eps);  // Simplified updating of x[index]
    }
}