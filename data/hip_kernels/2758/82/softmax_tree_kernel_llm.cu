#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device_new_api(float *input, int n, float temp, int stride, float *output)
{
    float sum = 0.0f;
    float largest = -INFINITY;
    
    // Use shared memory for reduction if within warp size (optimization)
    for (int i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = (val > largest) ? val : largest;
    }

    __shared__ float exp_sum[1024]; // Shared memory allocation for reduction
    
    for (int i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp);
        sum += e;
        output[i * stride] = e;
    }

    for (int i = 0; i < n; ++i) {
        output[i * stride] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;
    
    int s = id % spatial;
    id /= spatial;
    int g = id % groups;
    int b = id / groups;
    
    int goff = group_offset[g] * spatial;
    int boff = b * stride;
    
    softmax_device_new_api(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}