#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global thread index more efficiently by utilizing blockIdx.x
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Utilize stride increments to handle more data in a single kernel launch
    int stride = gridDim.x * blockDim.x;
    
    // Loop through data utilizing stride
    for (; i < N; i += stride)
    {
        Y[i * INCY] *= X[i * INCX];
    }
}