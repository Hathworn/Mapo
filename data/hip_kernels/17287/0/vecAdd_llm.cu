#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

int *a, *b;  // host data
int *c, *c2;  // results

__global__ void vecAdd(int *A, int *B, int *C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {  // Boundary check to avoid out-of-bounds access
        C[i] = A[i] + B[i];
    }
}