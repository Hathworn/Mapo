#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_imageSqrt(float * out, const float * in, const int width, const int height) {
    // Utilize shared memory for improved access speed
    __shared__ float localIn[blockDim.x * blockDim.y];
    
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int index = x + y * width;

    // Ensure thread index is within bounds
    if (x < width && y < height) {
        // Load data to shared memory and apply sqrt function
        localIn[threadIdx.y * blockDim.x + threadIdx.x] = in[index];
        __syncthreads();
        
        // Write output from shared memory to global memory
        out[index] = sqrtf(localIn[threadIdx.y * blockDim.x + threadIdx.x]);
    }
}