#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_colorRampHeatMapUnsat(uchar4 * colored, const float * vals, const int width, const int height, const float minVal, const float maxVal) {

    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Early exit if outside image bounds.
    if (x >= width || y >= height) return;

    const int index = x + y * width;
    uchar4 & imgVal = colored[index];

    // Handle NaN values.
    if (isnan(vals[index])) {
        imgVal = make_uchar4(0, 0, 0, 0);
        return;
    }

    // Normalize value in range 0 to 1.
    const float normVal = fmaxf(0, fminf((vals[index] - minVal) / (maxVal - minVal), 1));
    
    uchar3 a, b;

    // Assign colors based on normalized value range.
    if (normVal < 0.25f) {
        b = make_uchar3(32, 191, 139);
        a = make_uchar3(0x18, 0x62, 0x93);
    } else if (normVal < 0.5f) {
        b = make_uchar3(241, 232, 137);
        a = make_uchar3(32, 191, 139);
    } else if (normVal < 0.75f) {
        b = make_uchar3(198, 132, 63);
        a = make_uchar3(241, 232, 137);
    } else {
        b = make_uchar3(0xc0, 0x43, 0x36);
        a = make_uchar3(198, 132, 63);
    }

    // Calculate interpolation factor.
    const float t = (normVal < 1.0f) ? fmodf(normVal, 0.25f) * 4.0f : 1.0f;

    // Linearly interpolate between colors a and b.
    imgVal = make_uchar4(
                __float2int_rn((1.0f - t) * a.x + t * b.x),
                __float2int_rn((1.0f - t) * a.y + t * b.y),
                __float2int_rn((1.0f - t) * a.z + t * b.z),
                255);
}