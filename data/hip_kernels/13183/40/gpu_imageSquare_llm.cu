#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void gpu_imageSquare(float * out, const float * in, const int width, const int height) {
    // Pre-calculate commonly used values
    const int index = blockIdx.y * blockDim.y * width + blockIdx.x * blockDim.x + threadIdx.y * width + threadIdx.x;

    // Use grid-stride loop to cover all elements
    for (int idx = index; idx < width * height; idx += blockDim.x * gridDim.x * blockDim.y * gridDim.y) {
        // Ensure index is within the image bounds
        if (idx < width * height) {
            // Perform square operation
            out[idx] = in[idx] * in[idx];
        }
    }
}