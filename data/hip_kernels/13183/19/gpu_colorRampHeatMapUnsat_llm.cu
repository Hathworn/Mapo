#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_colorRampHeatMapUnsat(uchar3 * colored, const float * vals, const int width, const int height, const float minVal, const float maxVal) {

    // Calculate the x and y coordinates for the current thread
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Return if the thread is out of bounds
    if (x >= width || y >= height) {
        return;
    }

    const int index = x + y * width;
    uchar3 & imgVal = colored[index];

    // Check for NaN and set color to white
    if (isnan(vals[index])) {
        imgVal = make_uchar3(255, 255, 255);
        return;
    }

    const float normVal = fmaxf(0, fminf((vals[index] - minVal) / (maxVal - minVal), 1));
    const float t = normVal == 1.0f ? 1.0f : fmodf(normVal, 0.25f) * 4.0f;
    
    // Predefined color ramp values for different ranges
    uchar3 colors[4] = {
        make_uchar3(0x18, 0x62, 0x93),
        make_uchar3(32, 191, 139),
        make_uchar3(241, 232, 137),
        make_uchar3(198, 132, 63),
        make_uchar3(0xc0, 0x43, 0x36)
    };

    uchar3 a, b;
    if (normVal < 0.25f) {
        a = colors[0]; b = colors[1];
    } else if (normVal < 0.5f) {
        a = colors[1]; b = colors[2];
    } else if (normVal < 0.75f) {
        a = colors[2]; b = colors[3];
    } else {
        a = colors[3]; b = colors[4];
    }

    // Interpolate colors and assign to imgVal
    imgVal = make_uchar3((1 - t) * a.x + t * b.x,
                         (1 - t) * a.y + t * b.y,
                         (1 - t) * a.z + t * b.z);
}