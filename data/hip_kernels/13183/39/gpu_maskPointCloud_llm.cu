#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_maskPointCloud(float4* verts, const int width, const int height, const int* mask) {
    // Use 2D index calculation instead of separate x, y variables
    const int index = blockIdx.x * blockDim.x + threadIdx.x 
                    + (blockIdx.y * blockDim.y + threadIdx.y) * width;

    if (index >= width * height) return;

    // Avoid branch divergence with ternary operator
    verts[index].w = (mask[index] == 0) ? -1 : verts[index].w;
}