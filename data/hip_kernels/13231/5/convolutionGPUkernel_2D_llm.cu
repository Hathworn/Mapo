#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolutionGPUkernel_2D(int *M, int *mascara, int *resultado, int m, int n, int widthM) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within image boundaries
    if (col < n && row < m) {
        int p = 0;
        int start_col = col - (widthM / 2);
        int start_row = row - (widthM / 2);

        #pragma unroll  // Optimize with unrolling for better performance
        for (int i = 0; i < widthM; i++) {
            #pragma unroll
            for (int j = 0; j < widthM; j++) {
                int curRow = start_row + i;
                int curCol = start_col + j;
                
                // Update row-major index calculation logic
                if (curRow >= 0 && curRow < m && curCol >= 0 && curCol < n) {
                    p += M[curRow * n + curCol] * mascara[i * widthM + j];
                }
            }
        }
        
        // Assign computed value to the result matrix
        resultado[row * n + col] = p;
    }
}