#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelMultMat(int *a, int *b, int *c, int m) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < m && row < m) {
        int add = 0;
        
        // Reduce memory accesses by loading row elements once
        for (int i = 0; i < m; i++) {
            int a_element = a[row * m + i];
            int b_element = b[i * m + col];
            add += a_element * b_element;
        }
        
        c[row * m + col] = add;
    }
}