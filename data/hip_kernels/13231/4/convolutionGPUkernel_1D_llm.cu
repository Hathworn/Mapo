#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolutionGPUkernel_1D(int *h_n, int *h_mascara, int *h_r, int n, int mascara) {
    int mitadMascara = (mascara / 2);
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        int p = 0;
        int k = i - mitadMascara;

        #pragma unroll // Unroll the loop for better performance
        for (int j = 0; j < mascara; j++) {
            if (k < n && k >= 0) {
                p += h_n[k] * h_mascara[j];
            }
            k++;
        }

        h_r[i] = p; // Store result
    }
}