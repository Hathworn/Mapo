#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelMultMatTiled(float *d_M, float *d_N, float *d_P, int m, int n, int y) {

    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;
    float Pvalue = 0;

    // Loop unrolling for increased efficiency
    for(int i = 0; i < n / TILE_WIDTH; i++) {
        // Load tiles into shared memory with bounds check
        Mds[ty][tx] = (i * TILE_WIDTH + tx < n && row < m) ? d_M[row * n + i * TILE_WIDTH + tx] : 0.0f;
        Nds[ty][tx] = (i * TILE_WIDTH + ty < n && col < y) ? d_N[(i * TILE_WIDTH + ty) * y + col] : 0.0f;
        __syncthreads();

        // Unroll inner loop for better performance
        #pragma unroll
        for(int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += Mds[ty][k] * Nds[k][tx];
        }
        __syncthreads();
    }

    // Store the result in d_P with bounds check
    if(row < m && col < y) {
        d_P[row * y + col] = Pvalue;
    }
}