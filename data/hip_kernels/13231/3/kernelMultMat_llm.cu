#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelMultMat(double *d_a, double *d_b, double *d_c, int ROWS, int COL_A, int COL_B) {
    // Calculate global row and column indices
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check boundary conditions
    if (row < ROWS && col < COL_B) {
        double add = 0.0; // Accumulate result in a register
        // Loop unrolling (if beneficial) could be considered here
        for (int k = 0; k < COL_A; k++) {
            add += d_a[row * COL_A + k] * d_b[k * COL_B + col];
        }
        d_c[row * COL_B + col] = add; // Write result to global memory
    }
}