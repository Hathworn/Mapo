#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate the unique thread index
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;

    // Initialize the CURAND state
    hiprand_init(seed, tidx, 0, &state[tidx]);
}