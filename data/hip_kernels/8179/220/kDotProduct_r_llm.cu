#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f; // Initialize local sum

    // Loop unrolling for enhanced performance
    if (eidx + 3 * DP_BLOCKSIZE < numElements) {
        sum += a[eidx] * b[eidx];
        sum += a[eidx + DP_BLOCKSIZE] * b[eidx + DP_BLOCKSIZE];
        sum += a[eidx + 2 * DP_BLOCKSIZE] * b[eidx + 2 * DP_BLOCKSIZE];
        sum += a[eidx + 3 * DP_BLOCKSIZE] * b[eidx + 3 * DP_BLOCKSIZE];
    }

    // Write local sum to shared memory
    shmem[threadIdx.x] = sum;
    __syncthreads();
    
    // Reduce partial results in shared memory
    for (unsigned int s = DP_BLOCKSIZE / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            shmem[threadIdx.x] += shmem[threadIdx.x + s];
        }
        __syncthreads();
    }
    
    // Write final result to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}