#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use blockIdx and threadIdx to ensure only one thread performs the division
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        *accuracy /= N;
    }
}