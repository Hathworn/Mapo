#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    // Use shared memory for block-wise src fetching
    __shared__ float tile[1024]; // Assuming block size does not exceed 1024 threads
    
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Coalesced read from global to shared memory
        if (threadIdx.x < srcWidth * srcHeight) {
            tile[threadIdx.x] = src[srcY * srcWidth + srcX];
        }
        __syncthreads();

        // Write to global memory using data from shared memory
        tgt[i] = tile[(srcY % srcHeight) * srcWidth + (srcX % srcWidth)];
        __syncthreads();
    }
}