#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;
    
    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];
        
        // Use more efficient intrinsic function for log
        labelLogProbs[tx] = __logf(labelp);

        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            // Unroll loop for better performance
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp);
            }
            // Avoid division by zero, though it's theoretically unlikely
            correctProbs[tx] = (numMax > 0) ? (1.0f / float(numMax)) : 0.0f;
        }
    }
}