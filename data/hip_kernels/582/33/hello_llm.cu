#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Optimize printf by storing block index in a register
    int blockIndex = blockIdx.x;
    printf("Hello world! I'm a thread in block %d\n", blockIndex);
}