#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// http://en.wikipedia.org/wiki/Bitonic_sort

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    // Calculate the offset based on the stage and substage       
    for (int stage = 0; stage <= 5; stage++)
    {
        for (int substage = stage; substage >= 0; substage--)
        {
            int pairDistance = 1 << substage;  // Calculate pair distance    
            int blockDistance = 1 << stage;    // Calculate block distance  
            
            int idx =  tid ^ pairDistance;     // Determine comparison index using xor 
            float temp;

            // Ensure accessing valid indices with ordered element comparisons
            if (idx > tid)
            {
                if ((tid & blockDistance) == 0)
                {
                    if (sdata[tid] > sdata[idx])
                    {
                        temp = sdata[tid];
                        sdata[tid] = sdata[idx];
                        sdata[idx] = temp;
                    }
                }
                else
                {
                    if (sdata[tid] < sdata[idx])
                    {
                        temp = sdata[tid];
                        sdata[tid] = sdata[idx];
                        sdata[idx] = temp;
                    }
                }
            }
            __syncthreads();
        }
    }

    d_out[tid] = sdata[tid];
}