#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogram(const float* d_in, unsigned int* d_out, const float lumMin, const float lumRange, const size_t numBins, const size_t size)
{
    int abs_x = threadIdx.x + blockDim.x * blockIdx.x;

    // Check if the thread's index is within bounds
    if (abs_x >= size)
    {
        return;
    }

    // Calculate bin index
    int bin = (d_in[abs_x] - lumMin) / lumRange * numBins;

    // Use atomic operation to update histogram bin
    atomicAdd(&(d_out[bin]), 1);
}