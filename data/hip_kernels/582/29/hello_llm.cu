#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use blockDim.x instead of threadIdx.x for more threads across blocks
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello world! I'm thread %d\n", globalThreadId);
}