#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Cache commonly used values
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;

    // Ensure atomic operation with shared memory optimization
    extern __shared__ int s_bins[];
    atomicAdd(&(s_bins[myBin]), 1);
    __syncthreads(); // Synchronize threads within the block

    // Commit shared results back to global memory ensuring data consistency
    atomicAdd(&(d_bins[myBin]), s_bins[myBin]);
}