#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void exclusivePrefixAdd(unsigned int* d_in, unsigned int* d_out)
{
    // Belloch implementation
    // NOTE: This is set up specifically for 1 block of 1024 threads

    int thread_x = threadIdx.x;
    
    // Load data from global memory to shared memory to reduce global memory accesses
    __shared__ unsigned int temp[1024];
    temp[thread_x] = d_in[thread_x];
    __syncthreads();

    // First, do the reduce:
    for (unsigned int i = 2; i <= blockDim.x; i <<= 1)
    {
        if ((thread_x + 1) % i == 0)
        {
            temp[thread_x] += temp[thread_x - i / 2];
        }
        __syncthreads();
    }

    // Now do the downsweep part:
    if (thread_x == blockDim.x - 1)
    {
        temp[thread_x] = 0;
    }
    __syncthreads();

    for (unsigned int i = blockDim.x; i >= 2; i >>= 1)
    {
        if ((thread_x + 1) % i == 0)
        {
            unsigned int tmp = temp[thread_x - (i / 2)];
            // the "left" copy
            temp[thread_x - (i / 2)] = temp[thread_x];
            // and the "right" operation
            temp[thread_x] = tmp + temp[thread_x];
        }
        __syncthreads();
    }

    // Write back the result to the global memory
    d_out[thread_x] = temp[thread_x];
}