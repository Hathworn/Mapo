#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inclusivePrefixAdd(unsigned int* d_in, unsigned int* d_out)
{
    // Preprocess input into shared memory for better memory access pattern
    int abs_x = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_x = threadIdx.x;
    
    extern __shared__ unsigned int segment[];
    segment[thread_x] = d_in[abs_x];
    __syncthreads();

    // Hillis-Steele scan loop with shared memory optimization
    for (unsigned int i = 1; i < blockDim.x; i <<= 1)
    {
        unsigned int val = 0;
        if (thread_x >= i)
        {
            val = segment[thread_x - i];
        }
        __syncthreads(); // Ensure all threads have read before updating
        segment[thread_x] += val;
        __syncthreads(); // Ensure all updates complete before the next iteration
    }

    // Handle carry-over between blocks
    if (blockIdx.x > 0)
    {
        unsigned int carry = d_out[blockDim.x * (blockIdx.x - 1)];
        segment[thread_x] += carry;
    }

    d_out[abs_x] = segment[thread_x];
}