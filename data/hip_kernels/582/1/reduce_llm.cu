#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int* s) {
    // Load value into shared memory
    s[threadIdx.x] = p;

    // Ensure all values are written
    __syncthreads();

    // Perform reduction using warp shuffle
    unsigned int sum = p;
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
    }

    // Store the result in shared memory for the first thread
    if (threadIdx.x % warpSize == 0)
        s[threadIdx.x] = sum;

    // Sync threads and return reduction result for first thread in warp
    __syncthreads();
    return s[0];
}

__global__ void reduce(unsigned int* d_out_shared, const unsigned int* d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    if (t == 0) {
        *d_out_shared = sr;
    }
}