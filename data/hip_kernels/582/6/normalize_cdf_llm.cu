#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n) {
    // Calculate normalization constant once per block
    __shared__ float normalization_constant;
    if (threadIdx.x == 0) {
        normalization_constant = 1.f / d_input_cdf[n - 1];
    }
    __syncthreads(); // Ensure all threads have the normalization constant

    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_index_1d < n) {
        unsigned int input_value = d_input_cdf[global_index_1d];
        float output_value = input_value * normalization_constant;
        d_output_cdf[global_index_1d] = output_value;
    }
}