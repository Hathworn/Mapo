#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_reduce_kernel(float* d_out, float* d_in)
{
    extern __shared__ float s_data[]; // Allocate shared memory
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;
    
    // Load elements into shared memory
    s_data[tid] = d_in[myId];
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads(); // Ensure all operations at one stage are complete
    }

    // Write result for block to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = s_data[0];
    }
}