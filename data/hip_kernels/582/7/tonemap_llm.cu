#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tonemap( float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new, float  min_log_Y, float  max_log_Y, float  log_Y_range, int    num_bins, int    num_pixels_y, int    num_pixels_x )
{
    int nx = num_pixels_x;
    int2 image_index_2d = make_int2( ( blockIdx.x * blockDim.x ) + threadIdx.x, ( blockIdx.y * blockDim.y ) + threadIdx.y );
    int image_index_1d = ( nx * image_index_2d.y ) + image_index_2d.x;

    // Check if the current thread is within image boundaries
    if ( image_index_2d.x < num_pixels_x && image_index_2d.y < num_pixels_y )
    {
        // Use shared memory to reduce global memory access latency
        extern __shared__ float shared_mem[];
        float* shared_x = shared_mem;
        float* shared_y = shared_mem + blockDim.x * blockDim.y;
        float* shared_log_Y = shared_y + blockDim.x * blockDim.y;

        shared_x[threadIdx.x + threadIdx.y * blockDim.x] = d_x[image_index_1d];
        shared_y[threadIdx.x + threadIdx.y * blockDim.x] = d_y[image_index_1d];
        shared_log_Y[threadIdx.x + threadIdx.y * blockDim.x] = d_log_Y[image_index_1d];

        __syncthreads(); // Synchronize threads in the block

        float x = shared_x[threadIdx.x + threadIdx.y * blockDim.x];
        float y = shared_y[threadIdx.x + threadIdx.y * blockDim.x];
        float log_Y = shared_log_Y[threadIdx.x + threadIdx.y * blockDim.x];

        int bin_index = min( num_bins - 1, int( (num_bins * ( log_Y - min_log_Y ) ) / log_Y_range ) );
        float Y_new = d_cdf_norm[ bin_index ];

        // Calculate new color values
        float X_new = x * ( Y_new / y );
        float Z_new = ( 1 - x - y ) * ( Y_new / y );

        float r_new = ( X_new *  3.2406f ) + ( Y_new * -1.5372f ) + ( Z_new * -0.4986f );
        float g_new = ( X_new * -0.9689f ) + ( Y_new *  1.8758f ) + ( Z_new *  0.0415f );
        float b_new = ( X_new *  0.0557f ) + ( Y_new * -0.2040f ) + ( Z_new *  1.0570f );

        // Write results back to global memory
        d_r_new[ image_index_1d ] = r_new;
        d_g_new[ image_index_1d ] = g_new;
        d_b_new[ image_index_1d ] = b_new;
    }
}