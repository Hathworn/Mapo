#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shmem_reduce_kernel(float *d_out, const float *d_in)
{
    // Allocate shared memory
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // Load shared memory from global memory
    sdata[tid] = d_in[myId];
    __syncthreads(); // Synchronize to ensure the entire block is loaded

    // Unroll reduction loop for better performance
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Synchronize at each stage
    }

    // Reduction for the last warp without synchronization
    if (tid < 32) {
        volatile float *vsdata = sdata;
        vsdata[tid] += vsdata[tid + 32];
        vsdata[tid] += vsdata[tid + 16];
        vsdata[tid] += vsdata[tid + 8];
        vsdata[tid] += vsdata[tid + 4];
        vsdata[tid] += vsdata[tid + 2];
        vsdata[tid] += vsdata[tid + 1];
    }

    // Write result for this block back to global memory
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}