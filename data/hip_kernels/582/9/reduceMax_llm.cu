#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceMax(const float* d_in, float* d_out) {
    extern __shared__ float sdata[];
    int abs_x = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_x = threadIdx.x;

    // Load shared memory in bounds
    sdata[thread_x] = d_in[abs_x];
    __syncthreads();

    // Reduce block data to a single maximum value
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (thread_x < s) {
            sdata[thread_x] = max(sdata[thread_x], sdata[thread_x + s]);
        }
        __syncthreads();
    }

    // Return result at the 0th thread of every block
    if (thread_x == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}