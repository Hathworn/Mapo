#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gGetValueByKey(float* d_in, float* d_out, int* indeces, int n) {
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid < n) {
        // Load index value only once into a register
        int index = __ldg(&indeces[tid]); 
        d_out[tid] = __ldg(&d_in[index]); // Use read-only cache for global memory access
    }
}