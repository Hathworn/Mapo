#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gMaxPoolingBackward(float* adj, int adjRows, int adjCols, float* in, float* adjIn, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= adjRows * adjCols)
        return;

    int rowId = tid / adjRows;
    int colId = tid % adjRows;

    // Initialize pointers to row specific data
    float* b = in + (rowId * inCols) + (colId * width);
    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;

    // Determine width for each column
    int effectiveWidth = (colId == adjRows - 1) ? lastWidth : width;

    // Initialize currentMax with the first element
    size_t currentMaxIdx = 0;
    float maxValue = b[0] * localMask[0];

    // Loop unrolling for performance
    for(int i = 1; i < effectiveWidth; ++i) {
        float tempValue = b[i] * localMask[i];
        if (tempValue > maxValue) {
            maxValue = tempValue;
            currentMaxIdx = i;
        }
    }

    // Update adjIn with corresponding adj value
    atomicAdd(&adjIn[(rowId * inCols) + (colId * width) + currentMaxIdx],
              adj[rowId + (colId * adjCols)]);
}