#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gMaxPoolingForward(float* out, int outRows, int outCols, float* in, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if (tid >= outRows * outCols)
        return;

    int rowId = tid / outCols; // Fixed: Correct column indexing
    int colId = tid % outCols; // Fixed: Correct row indexing

    if (colId == outRows - 1) {
        width = lastWidth;
    }

    float* b = in + (rowId * inCols) + (colId * width);
    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;

    float currentMax = -INFINITY; // Optimization: Initialize with negative infinity
    for (int i = 0; i < width; ++i) {
        float val = b[i] * localMask[i];
        if (val > currentMax) {
            currentMax = val;
        }
    }

    out[tid] = currentMax; // Fixed: Use `tid` to index output directly
}