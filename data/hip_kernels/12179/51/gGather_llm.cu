#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gGather(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Check bounds first to exit early
    if (idx < sparseSize) { 
        int adjustedIndex = sparseIndices[idx] + offset;
        
        // Ensure adjusted index is within valid range
        if (adjustedIndex >= 0 && adjustedIndex < denseSize) { 
            sparseData[idx] = denseData[adjustedIndex];
        }
    }
}