#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gScatterUpdate(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use shared memory for improved memory access efficiency
    __shared__ int sharedSparseIndices[256];  // Assuming max blockDim.x of 256, adjust as necessary
    __shared__ float sharedSparseData[256];
    
    if (idx < sparseSize) {
        sharedSparseIndices[threadIdx.x] = sparseIndices[idx];
        sharedSparseData[threadIdx.x] = sparseData[idx];
    }
    
    __syncthreads();  // Ensure shared memory is populated

    if(idx < sparseSize) {
        int index = sharedSparseIndices[threadIdx.x];
        if(index >= -offset && index + offset < denseSize) {
            denseData[index + offset] = sharedSparseData[threadIdx.x];
        }
    }
}