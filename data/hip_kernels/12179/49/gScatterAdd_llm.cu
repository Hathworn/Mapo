#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gScatterAdd(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    // Calculate thread's unique global index
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Use stride loop to handle out-of-bound indices and improve ILP
    for (int i = idx; i < sparseSize; i += gridDim.x * blockDim.x) {
        // Check bounds before updating to ensure safety
        if (sparseIndices[i] >= -offset && sparseIndices[i] + offset < denseSize) {
            atomicAdd(&denseData[sparseIndices[i] + offset], sparseData[i]);
        }
    }
}