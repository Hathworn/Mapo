#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel by using grid-stride loop
__global__ void gSetSparse(float* out, const size_t* indices, const float* values, int length) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < length; i += stride) {
        out[indices[i]] = values[i];
    }
}