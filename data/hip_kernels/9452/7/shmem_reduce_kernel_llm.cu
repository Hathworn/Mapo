#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shmem_reduce_kernel(float * d_out, const float * d_in)
{
    // Shared data allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // Load shared memory from global memory
    sdata[tid] = d_in[myId];
    __syncthreads();  // Make sure entire block is loaded

    // Perform reduction in shared memory
    // Use warp-synchronous to reduce synchronization overhead
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();  // Ensure all adds at one stage are done
    }

    // Only thread 0 writes the result back to global memory for this block
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}