#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cube(float *d_out, float *d_in) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x; // Calculate global thread index
    float f = d_in[idx];
    d_out[idx] = f * f * f;
}