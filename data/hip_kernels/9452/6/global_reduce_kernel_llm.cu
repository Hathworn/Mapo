#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
    // Using shared memory for intra-block reduction
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // Load input into shared memory
    sdata[tid] = d_in[myId];
    __syncthreads();

    // Do reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Ensure all threads complete current stage
    }

    // Only thread 0 writes result for this block back to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}