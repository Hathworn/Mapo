#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void remove_redness_from_coordinates( const unsigned int* d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, int num_coordinates, int num_pixels_y, int num_pixels_x, int template_half_height, int template_half_width )
{
    int nx = num_pixels_x;
    int ny = num_pixels_y;
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;
    int imgSize = nx * ny;

    if (global_index_1d < num_coordinates)
    {
        unsigned int image_index_1d = d_coordinates[imgSize - global_index_1d - 1];
        ushort2 image_index_2d = make_ushort2(image_index_1d % nx, image_index_1d / nx);

        // Iterate over the template area in the image
        int startX = max(0, image_index_2d.x - template_half_width);
        int endX = min(nx - 1, image_index_2d.x + template_half_width);
        int startY = max(0, image_index_2d.y - template_half_height);
        int endY = min(ny - 1, image_index_2d.y + template_half_height);

        for (int y = startY; y <= endY; y++)
        {
            for (int x = startX; x <= endX; x++)
            {
                int image_offset_index_1d_clamped = y * nx + x;

                unsigned char g_value = d_g[image_offset_index_1d_clamped];
                unsigned char b_value = d_b[image_offset_index_1d_clamped];

                unsigned int gb_average = (g_value + b_value) >> 1; // Bitwise shift for division by 2

                d_r_output[image_offset_index_1d_clamped] = (unsigned char)gb_average;
            }
        }
    }
}