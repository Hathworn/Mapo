#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tonemap(float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new, float min_log_Y, float max_log_Y, float log_Y_range, int num_bins, int num_pixels_y, int num_pixels_x)
{
    int nx = num_pixels_x;
    int ny = num_pixels_y;
    int2 image_index_2d = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    
    if (image_index_2d.x < nx && image_index_2d.y < ny) {
        int image_index_1d = nx * image_index_2d.y + image_index_2d.x;

        // Directly access memory to reduce variable count
        float x = __ldg(&d_x[image_index_1d]);
        float y = __ldg(&d_y[image_index_1d]);
        float log_Y = __ldg(&d_log_Y[image_index_1d]);

        int bin_index = min(num_bins - 1, int(num_bins * (log_Y - min_log_Y) / log_Y_range));
        float Y_new = __ldg(&d_cdf_norm[bin_index]);

        // Precompute common subexpression to reduce arithmetic operations
        float inv_y = 1.0f / y;
        float Y_div_y = Y_new * inv_y;
        float X_new = x * Y_div_y;
        float Z_new = (1.0f - x - y) * Y_div_y;

        // Optimize color transformation by using fewer operations
        float r_new = fmaf(X_new, 3.2406f, fmaf(Y_new, -1.5372f, Z_new * -0.4986f));
        float g_new = fmaf(X_new, -0.9689f, fmaf(Y_new, 1.8758f, Z_new * 0.0415f));
        float b_new = fmaf(X_new, 0.0557f, fmaf(Y_new, -0.2040f, Z_new * 1.0570f));

        d_r_new[image_index_1d] = r_new;
        d_g_new[image_index_1d] = g_new;
        d_b_new[image_index_1d] = b_new;
    }
}