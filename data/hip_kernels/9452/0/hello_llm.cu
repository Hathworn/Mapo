#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void hello()
{
    // Use shared memory to reduce redundant printf call
    __shared__ int block_id;
    if (threadIdx.x == 0) {
        block_id = blockIdx.x;
    }
    __syncthreads(); // Ensure all threads see the updated block_id

    printf("Hello world! I'm a thread in block %d\n", block_id);
}