#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Block index Printf (Rodrigo)

#define NUM_BLOCKS 16
#define BLOCK_WIDTH 1

__global__ void hello()
{
    int threadId = threadIdx.x; // Cache thread index in register
    printf("Hello world! I'm thread %d\n", threadId);
}