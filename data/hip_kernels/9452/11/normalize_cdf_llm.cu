#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    // Compute normalization constant once for all threads
    const float normalization_constant = 1.f / d_input_cdf[n - 1];
    
    // Compute global index
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check boundary condition
    if (global_index_1d < n)
    {
        // Load input data directly into registers for faster access
        unsigned int input_value = d_input_cdf[global_index_1d];
        float output_value = input_value * normalization_constant;
        
        // Store result
        d_output_cdf[global_index_1d] = output_value;
    }
}