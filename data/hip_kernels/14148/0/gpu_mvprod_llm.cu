#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// number of threads in a block (must be multiple of 32)
#define NUMTHREADS 128

// the maximum size of the grid
#define GRIDSIZE 65535

// Helper macro to check CUDA return values
__global__ void gpu_mvprod(double *x, double *y, double alpha, int NumUp, int NumDown, int dim, double *Umat, double *Down_data,unsigned int *Down_ind, int size_Down, double *Up_data, unsigned int *Up_ind, int size_Up, int rows_shared)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index < dim)
    {
        double result = Umat[index] * x[index];

        int sv = index / NumDown;
        int id = index % NumDown;

        extern __shared__ double shared[];

        unsigned int *shared_ind = (unsigned int *) &shared[size_Up * rows_shared];

        int s_sv = blockIdx.x * blockDim.x / NumDown;

        // Use coalesced access pattern for shared memory initialization
        if(threadIdx.x < rows_shared && (s_sv + threadIdx.x) < NumUp)
        {
            for(int i = 0; i < size_Up; i++)
            {
                shared[i * rows_shared + threadIdx.x] = Up_data[s_sv + threadIdx.x + i * NumUp];
                shared_ind[i * rows_shared + threadIdx.x] = Up_ind[s_sv + threadIdx.x + i * NumUp];
            }
        }

        __syncthreads();

        for(int i = 0; i < size_Up; i++)
        {
            result += shared[sv - s_sv + i * rows_shared] * x[id + NumDown * shared_ind[sv - s_sv + i * rows_shared]];
        }

        for(int i = 0; i < size_Down; i++)
        {
            result += Down_data[id + i * NumDown] * x[sv * NumDown + Down_ind[id + i * NumDown]];
        }

        y[index] = alpha * y[index] + result;
    }
}