#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_kernel_initRND(unsigned long seed, hiprandState *States)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified id calculation
    int pixel = id; // Direct mapping to id
    hiprand_init(seed, pixel, 0, &States[id]); // Initialize random state
}