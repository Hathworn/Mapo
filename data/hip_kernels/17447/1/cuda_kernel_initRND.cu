#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_kernel_initRND(unsigned long seed, hiprandState *States)
{
int tid = threadIdx.x;
int bid = blockIdx.x;

int id    = bid*RND_BLOCK_SIZE + tid;
int pixel = bid*RND_BLOCK_SIZE + tid;

hiprand_init(seed, pixel, 0, &States[id]);
}