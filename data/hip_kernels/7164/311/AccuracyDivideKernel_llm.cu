#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Divide only if thread 0 executes (single execution for division)
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}