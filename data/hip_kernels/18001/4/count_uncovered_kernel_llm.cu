#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void count_uncovered_kernel(size_t batch_size, size_t num_nodes, uint32_t *d_rrr_index, uint32_t *d_rrr_sets, char *d_mask, uint32_t *d_counters) {
    int pos = threadIdx.x + blockDim.x * blockIdx.x;

    if (pos < batch_size) {
        uint32_t set = d_rrr_index[pos];
        // Improved memory access, load once
        char mask_set = d_mask[set];
        if (mask_set != 1) {
            uint32_t v = d_rrr_sets[pos];
            // Using atomic to ensure correct concurrent updates
            atomicAdd(d_counters + v, 1);
        }
    }
}