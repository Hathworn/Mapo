#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_vectors(uint32_t * src, uint32_t * dst, size_t N) {
    size_t pos = threadIdx.x + blockDim.x * blockIdx.x;
    if (pos < N) {
        uint32_t value = src[pos];
        if (value != 0) {
            atomicAdd(&dst[pos], value); // Directly add when non-zero
        }
    }
}