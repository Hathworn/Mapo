#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_mask_kernel(size_t batch_size, uint32_t *d_rrr_index, uint32_t *d_rrr_sets, char *d_mask, uint32_t last_seed) {
    size_t pos = threadIdx.x + blockDim.x * blockIdx.x;

    // Ensure valid thread accesses with early return
    if (pos >= batch_size) return;

    // Combine condition checks for more efficient execution
    if (d_rrr_sets[pos] == last_seed) {
        uint32_t set = d_rrr_index[pos];
        d_mask[set] = 1;
    }
}