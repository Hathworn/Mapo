#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;
  
    // Unroll loop for better performance
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();
    
    // Use efficient reduction to compute inner products
    if (idx < 32) {
        for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
            if (idx < offset) {
                reduction_buffer[idx] += reduction_buffer[idx + offset];
            }
            __syncthreads();
        }
    }
    
    // Reuse shared memory to store the final sum
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}