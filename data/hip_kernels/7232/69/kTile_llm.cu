#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate the thread's unique index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the total number of threads
    const int numThreads = blockDim.x * gridDim.x;

    // Calculate the total number of elements
    const unsigned int numEls = tgtWidth * tgtHeight;

    // Process multiple elements per thread in each iteration
    for (uint i = idx; i < numEls; i += numThreads) {
        // Calculate the target's x and y coordinates
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;

        // Calculate the corresponding source coordinates (tiled mapping)
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Assign the source value to the target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}