#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;

    // Unroll the loop for better performance
    if (eidx < numElements) {
        for (uint i = eidx; i < numElements; i += gridDim.x * DP_BLOCKSIZE) {
            shmem[threadIdx.x] += a[i] * b[i];
        }
    }
    __syncthreads();

    // Unroll reduction steps
    if (threadIdx.x < 256) shmem[threadIdx.x] += shmem[threadIdx.x + 256];
    __syncthreads();
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();

    // Use warp-level shuffle for efficiency
    if (threadIdx.x < 32) {
        float val = shmem[threadIdx.x];
        val += __shfl_down(val, 16);
        val += __shfl_down(val, 8);
        val += __shfl_down(val, 4);
        val += __shfl_down(val, 2);
        val += __shfl_down(val, 1);

        if (threadIdx.x == 0) {
            target[blockIdx.x] = val;
        }
    }
}