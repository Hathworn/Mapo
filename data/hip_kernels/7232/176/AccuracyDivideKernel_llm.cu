#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load accuracy into register
    float acc = *accuracy;
    // Perform division using register
    acc /= N;
    // Store the result back into global memory
    *accuracy = acc;
}