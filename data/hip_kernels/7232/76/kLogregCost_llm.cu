#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = static_cast<int>(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        if (labelp != maxp) {
            correctProbs[tx] = 0.0f;
        } else {
            int numMax = 0;
            // Optimize the loop using unrolling for potential performance gain.
            for (int i = 0; i < numOut; i += 4) {
                numMax += (i < numOut && probs[i * numCases + tx] == maxp);
                numMax += ((i + 1) < numOut && probs[(i + 1) * numCases + tx] == maxp);
                numMax += ((i + 2) < numOut && probs[(i + 2) * numCases + tx] == maxp);
                numMax += ((i + 3) < numOut && probs[(i + 3) * numCases + tx] == maxp);
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}