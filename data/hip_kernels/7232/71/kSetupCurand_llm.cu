#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    // Use a unique seed to avoid repeated sequences across threads
    hiprand_init(seed + tidx, 0, 0, &state[tidx]);
}