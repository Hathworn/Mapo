#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void checkIndex() {
    // Optimized by merging printf statement into one line
    printf("Thread: (%d, %d, %d) Block: (%d, %d, %d) Block Dim: (%d, %d, %d) Grid Dim: (%d, %d, %d)\n", 
           threadIdx.x, threadIdx.y, threadIdx.z, 
           blockIdx.x, blockIdx.y, blockIdx.z, 
           blockDim.x, blockDim.y, blockDim.z, 
           gridDim.x, gridDim.y, gridDim.z);
}