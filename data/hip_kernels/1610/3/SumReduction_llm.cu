#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SumReduction(int* input, int n)
{
    // Handle to thread block group
    extern __shared__ int sm[];

    // Load shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    sm[tid] = (i < n) ? input[i] : 0;
    
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) // Optimize loop range and step
    {
        if (tid < s)
        {
            sm[tid] += sm[tid + s];
        }
        
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0)
    {
        input[blockIdx.x] = sm[0];
    }
}