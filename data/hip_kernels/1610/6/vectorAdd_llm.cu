#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(int numElements, float *x, float *y)
{
    // Compute global index once
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure memory coalescing and avoid branching
    if (i < numElements)
    {
        y[i] += x[i];
    }
}