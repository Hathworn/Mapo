#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    // Calculate global index and stride to handle more elements in fewer blocks
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop over elements with a step size of stride
    for (; i < numElements; i += stride)
    {
        C[i] = A[i] + B[i];
    }
}