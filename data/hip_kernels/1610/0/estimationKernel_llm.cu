#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PI 3.14159265358979323846

#define N 10000  // data size
#define ES 10000 // estimation size
#define HS 20    // histogram size

__global__ void estimationKernel(float* data, size_t n, float* kernelEstimation, size_t es, float dx, float h)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Precompute constant factors outside the loop
    float h_inv = 1.0f / h;
    float norm_factor = 1.0f / (n * h * sqrtf(2 * PI));

    for (int i = index; i < es; i += stride)
    {
        float di = dx * i; // current OX axis position
        float sum = 0.0f;  // initialize sum for current kernel estimation

        for (int j = 0; j < n; j++)
        {
            // Use precomputed h_inv for division
            float diff = di - data[j];
            float power = -0.5f * diff * diff * h_inv * h_inv;
            sum += expf(power);
        }
        
        // Apply normalization factor
        kernelEstimation[i] = sum * norm_factor;
    }
}