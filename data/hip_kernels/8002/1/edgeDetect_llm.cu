#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void edgeDetect(unsigned char* device_input_data, unsigned char* device_output_data, int height, int width) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    //check bounds
    if (x < 1 || x > width - 2 || y < 1 || y > height - 2)
        return;

    //for horizontal lines
    const int fmat_x[3][3] = {
        {-1, 0, 1},
        {-2, 0, 2},
        {-1, 0, 1}
    };
    // for vertical lines
    const int fmat_y[3][3]  = {
        {-1, -2, -1},
        {0,   0,  0},
        {1,   2,  1}
    };

    double G_x = 0;
    double G_y = 0;

    //go through rows and cols
    for (int i = -1; i <= 1; ++i) {
        for (int j = -1; j <= 1; ++j) {
            G_x += (double)(fmat_x[i + 1][j + 1] * device_input_data[(y + i) * width + (x + j)]);
            G_y += (double)(fmat_y[i + 1][j + 1] * device_input_data[(y + i) * width + (x + j)]);
        }
    }

    int G = sqrt(G_x * G_x + G_y * G_y);

    // Clamp the value to [0, 255]
    if (G > 255)
        G = 255;

    device_output_data[y * width + x] = G;
}