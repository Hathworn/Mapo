#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//macro to check return value of the cuda runtime call and exits
//if call failed
__global__ void anyMethod(unsigned char* buff , unsigned char* buffer_out , int w , int h)
{
    // Calculate the column index of the Px being processed
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the row index of the Px being processed
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int width = w, height = h;

    if (x < width && y < height && x > 0 && x < width - 1 && y > 0 && y < height - 1) 
    {
        int hx = -buff[width*(y-1) + (x-1)] + buff[width*(y-1)+(x+1)]
                 -2*buff[width*(y)+(x-1)] + 2*buff[width*(y)+(x+1)]
                 -buff[width*(y+1)+(x-1)] + buff[width*(y+1)+(x+1)];

        int vx = buff[width*(y-1)+(x-1)] + 2*buff[width*(y-1)+(x+1)] + buff[width*(y-1)+(x+1)]
                 - buff[width*(y+1)+(x-1)] - 2*buff[width*(y+1)+(x)] - buff[width*(y+1)+(x+1)];
        // Adjust intensity for effect
        hx = hx * 4;
        vx = vx / 5;

        int val = (int)sqrtf((float)(hx) * (float)(hx) + (float)(vx) * (float)(vx));

        buffer_out[y * width + x] = (unsigned char) val;
    }
}