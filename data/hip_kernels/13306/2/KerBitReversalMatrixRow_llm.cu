#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_DIM_X 16
#define BLOCK_DIM_Y 32

#define N 16

__global__ static void KerBitReversalMatrixRow(float *d_lpDstRe, float *d_lpDstIm, float *d_lpSrcRe, float *d_lpSrcIm, int width, int log2x)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Calculate index only for valid x
    if (x < (1 << log2x))
    {
        int index = __brev(x) >> (32 - log2x); // Use __brev to reverse bits of x
        if (x >= index)
        {
            // Compute memory addresses only once
            int idx = width * y + x;
            int jdx = width * y + index;

            // Use float type directly since data is float
            float tmpRe = d_lpDstRe[idx];
            float tmpIm = d_lpDstIm[idx];

            d_lpDstRe[idx] = d_lpSrcRe[jdx];
            d_lpDstIm[idx] = d_lpSrcIm[jdx];

            d_lpDstRe[jdx] = tmpRe;
            d_lpDstIm[jdx] = tmpIm;
        }
    }
}