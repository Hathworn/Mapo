#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_standarization(float *data, int rows, int columns) {
    int total_threads_count = blockDim.x * gridDim.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = tid + 1; i < columns; i += total_threads_count) {
        float amo = 0.0f, var = 0.0f;
        
        // Calculate sum and mean
        for (int j = 0; j < rows; ++j) {
            amo += data[j * columns + i];
        }
        float ave = amo / float(rows);

        // Calculate variance
        for (int j = 0; j < rows; ++j) {
            float factor = data[j * columns + i] - ave;
            var += (factor * factor);
        }

        // Check for zero variance
        if (var == 0.0f) {
            for (int j = 0; j < rows; j++) {
                data[j * columns + i] /= 255.0f;
            }
            continue;
        }

        float sd_reciprocal = 1.0f / sqrtf(var);

        // Standardize data
        for (int j = 0; j < rows; j++) {
            data[j * columns + i] = (data[j * columns + i] - ave) * sd_reciprocal;
        }
    }
}