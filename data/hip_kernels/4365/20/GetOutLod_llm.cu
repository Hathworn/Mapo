#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Load-balancing by assigning tasks to threads and 
   using shared memory for often-accessed data */
__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod, const size_t lod_len, size_t* out_lod0) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensuring index is within bounds
    if (index < lod_len) {
        // Use local index variable for reduced global memory access
        size_t in_lod_val = in_lod[index];
        out_lod0[index] = in_lod_val - num_erased[in_lod_val];
    }
}