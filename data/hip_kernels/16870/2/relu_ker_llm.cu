#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_ker(float* src, float* dst, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        // Use fmaxf to apply ReLU in a compact form
        dst[i] = fmaxf(0.0f, src[i]);
    }
}