#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function with improved memory access pattern
__global__ void debug_ker(float* ptr, int addr) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate thread index
    if (i == 0) { // Ensure only one thread prints to reduce contention
        printf("%d %f\n", addr, ptr[addr]); // Print the value at the specified address
    }
}