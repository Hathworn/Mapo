#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuExp(int n, float *a, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        // Use fast math intrinsic for exponentiation
        result[i] = __expf(a[i]);
    }
}