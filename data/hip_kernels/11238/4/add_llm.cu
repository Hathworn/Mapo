#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void add(int n, float *a, float *b, float *sum)
{
    // Calculate index using grid-stride loop for better parallel efficiency
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
    {
        sum[i] = a[i] + b[i];
    }
}