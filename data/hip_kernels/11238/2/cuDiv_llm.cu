#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"

__global__ void cuDiv(int n, float *a, float *b, float *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid stride loop for better parallelism
    for (; i < n; i += blockDim.x * gridDim.x) 
    {
        result[i] = a[i] / b[i];
    }
}