#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuMult(int n, float *a, float *b, float *result)
{
    // Use shared memory to improve memory access efficiency
    __shared__ float a_shared[256];
    __shared__ float b_shared[256];

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Only load data within bounds
    if (i < n)
    {
        a_shared[threadIdx.x] = a[i];
        b_shared[threadIdx.x] = b[i];
        __syncthreads();

        result[i] = a_shared[threadIdx.x] * b_shared[threadIdx.x];
    }
}