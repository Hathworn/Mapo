#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuAdd(int n, float *a, float *b, float *result)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure that all threads can be utilized
    int stride = gridDim.x * blockDim.x;

    // Process each element in strides to cover all data
    for (int idx = i; idx < n; idx += stride)
    {
        result[idx] = a[idx] + b[idx];
    }
}