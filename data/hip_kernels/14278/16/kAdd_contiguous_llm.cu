#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate thread's global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process multiple elements in each thread to increase throughput
    for (unsigned int i = idx; i < numEls; i += blockDim.x * gridDim.x) {
        dest[i] = a[i] + b[i];
    }
}