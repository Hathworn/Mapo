#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C"
__global__ void invert(uchar4* data, int w, int h)
{
    // Using 2D index calculation for better memory coalescing
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    if (x < w && y < h)
    {
        int index = y * w + x;        // Compute global thread index

        // Read pixel data
        uchar4 pixel = data[index];

        // Invert colors
        pixel.x = 255 - pixel.x;
        pixel.y = 255 - pixel.y;
        pixel.z = 255 - pixel.z;
        pixel.w = 255 - pixel.w;

        // Write back the inverted pixel
        data[index] = pixel;
    }
}