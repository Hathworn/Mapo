#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blockReduction(double* dN_pTdpTdphidy_d, int final_spectrum_size, int blocks_ker1)
{
    long idx = threadIdx.x + blockDim.x * blockIdx.x;

    // Launch threads only if within bounds
    if (idx < final_spectrum_size)
    {
        // This condition is redundant; optimization by using a simple loop in case of blocks_ker1 > 1
        for (int i = 1; i < blocks_ker1; ++i)
        {
            // Reduced memory access by calculating offset once
            int offset = idx + i * final_spectrum_size;

            // Optimize: Use warp shuffle or atomic addition if applicable for large data
            dN_pTdpTdphidy_d[idx] += dN_pTdpTdphidy_d[offset];

            // Check NaN and report
            if (isnan(dN_pTdpTdphidy_d[idx])) printf("found dN_pTdpTdphidy_d nan \n");
        }
    }
}