#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod, const size_t lod_len, size_t* out_lod0) {
    // Get the unique thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we do not read/write out of bounds
    if (index < lod_len) {
        size_t in_lod_val = in_lod[index];
        out_lod0[index] = in_lod_val - num_erased[in_lod_val];
    }
}