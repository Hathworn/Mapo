#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* __restrict__ accuracy) {
    // Use atomic operation to ensure thread safety
    atomicExch(accuracy, *accuracy / N);
}