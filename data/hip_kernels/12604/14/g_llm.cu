#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int d(void) { return 8; }

// Simplified to remove empty kernel function
__global__ void g(void) {
    // Empty kernel intentionally
}