#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel for Gaussian Elimination
__global__ void gauss_elimination_cuda(float *a_d, float *b_d ,int size) {
    int idx = threadIdx.x;
    int idy = threadIdx.y;

    // Utilize shared memory to optimize data access
    __shared__ float temp[40][40];
    temp[idy][idx] = a_d[(idy * (size+1)) + idx];
    __syncthreads();

    // Perform Gaussian elimination
    for (int column = 0; column < size - 1; column++) {
        if (idy > column && idx >= column) {
            // Optimize by reducing redundant calculation
            float multiplier = temp[column][column] / temp[idy][column];
            float t = temp[column][idx] - multiplier * temp[idy][idx];
            __syncthreads();
            temp[idy][idx] = t;
        }
        __syncthreads();
    }

    // Write results back to global memory
    b_d[idy * (size + 1) + idx] = temp[idy][idx];
}