#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define maximum size for shared memory
#define MAXSIZE 1024 

__global__ void gauss_elimination_cuda_new(float *a_d, float *b_d, int size) {
    int idy = threadIdx.x;

    // Use dynamic shared memory for flexibility
    extern __shared__ float temp[];

    // Each thread loads its row from global to shared memory
    for (int i = 0; i < size+1; i++) {
        temp[idy * (size+1) + i] = a_d[(idy * (size+1)) + i];
    }
    __syncthreads();

    // Perform Gaussian elimination
    for (int i = 1; i < size; i++) {
        if (idy >= i) {
            float factor = temp[idy * (size+1) + (i-1)] / temp[(i-1) * (size+1) + (i-1)];
            
            // Optimize loop to avoid boundary checking
            for (int j = i-1; j < size+1; j++) {
                temp[idy * (size+1) + j] -= temp[(i-1) * (size+1) + j] * factor;
            }
        }
        __syncthreads();
    }

    // Each thread writes its row back to global memory
    for (int i = 0; i < size+1; i++) {
        b_d[idy * (size+1) + i] = temp[idy * (size+1) + i];
    }
}