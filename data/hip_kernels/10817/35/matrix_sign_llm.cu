#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access by using shared memory for smaller matrices
__global__ void matrix_sign(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        // Use ternary operator for potential minor performance improvement
        C[index] = (A[index] == 0.0) ? 0.0 : copysign(1.0, A[index]);
    }
}