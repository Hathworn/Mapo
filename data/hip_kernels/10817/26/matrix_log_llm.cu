#include "hip/hip_runtime.h"
```c++
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to compute the natural logarithm of each element
extern "C"
__global__ void matrix_log(double *A, double *C, unsigned int size) {
    // Use built-in function to compute global index
    unsigned int index = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;
    
    // Ensure index is within array bounds
    if (index < size) {
        // Calculate logarithm of the element at index
        C[index] = log(A[index]);
    }
}