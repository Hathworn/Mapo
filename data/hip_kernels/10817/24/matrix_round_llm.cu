#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/
extern "C"
__global__ void matrix_round(double *A, double *C, unsigned int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Using shared memory for better caching if applicable
    __shared__ double sharedA[256];  // Adjust size depending on the GPU's shared memory available
    if (index < size) {
        sharedA[threadIdx.x] = A[index];
        __syncthreads(); // Ensure all threads have written to shared memory

        // Perform operation with cached shared memory
        C[index] = (double)llround(sharedA[threadIdx.x]);
    }
}