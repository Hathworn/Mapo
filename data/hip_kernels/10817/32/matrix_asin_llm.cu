#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/
extern "C"
__global__ void matrix_asin(double *A, double *C, unsigned int size) {
    // Calculate the global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use loop unrolling for better performance
    if (index < size) {
        C[index] = asin(A[index]);
    }
}