#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**********************************
When updating a kernel or adding a new one,
please compile the ptx file and commit it:
nvcc -ptx -arch=sm_30 SystemML.cu
***********************************/

/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/
extern "C"

__global__ void matrix_sin(double *A, double *C, unsigned int size) {
    // Calculate index for this thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory cache to optimize sin calculations if needed
    if (index < size) {
        C[index] = __sin(A[index]); // Use __sin for potential CUDA math lib optimizations
    }
}