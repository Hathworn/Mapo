#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel for better performance
extern "C"
__global__ void compare_and_set(double* A, double* ret, int rlen, int clen, double compareVal, double tol, double ifEqualsVal, double ifLessThanVal, double ifGreaterThanVal) {
    // Calculate global thread index
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (ix < rlen && iy < clen) {
        int index = ix * clen + iy;

        // Calculate the difference once to avoid repetitive calculation
        double diff = A[index] - compareVal;

        // Use ternary operator for concise conditional assignment
        ret[index] = (abs(diff) < tol) ? ifEqualsVal : (diff < 0) ? ifLessThanVal : ifGreaterThanVal;
    }
}