#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/

extern "C"
__global__ void bias_multiply(double* input, double* bias, double* ret, int rlen, int clen, int PQ) {
    // Calculate unique thread index for mapping
    int index = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * clen;
    
    // Ensure unique index is within bounds
    if(index < rlen * clen) {
        int iy = index % clen; // Calculate column based on index
        int biasIndex = iy / PQ; // Find bias index
        ret[index] = input[index] * bias[biasIndex];
    }
}