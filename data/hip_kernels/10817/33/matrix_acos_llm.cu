#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/
extern "C"
__global__ void matrix_acos(double *A, double *C, unsigned int size) {
    // Calculate the linear index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for faster access within a block (optional, based on use case)
    // Perform acos computation only for valid indices
    if (index < size) {
        C[index] = acos(A[index]);
    }
}