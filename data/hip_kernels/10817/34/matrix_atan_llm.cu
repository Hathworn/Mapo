#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/
extern "C"
__global__ void matrix_atan(double *A, double *C, unsigned int size) {
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop unrolling to maximize throughput
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < size; i += stride) {
        C[i] = atan(A[i]);
    }
}