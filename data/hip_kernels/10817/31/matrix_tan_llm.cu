#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/
extern "C"
__global__ void matrix_tan(double *A, double *C, unsigned int size) {
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory to cache input data (optional, example purpose only)
    __shared__ double sharedA[256];
    
    if (index < size) {
        // Load data into shared memory from global memory
        sharedA[threadIdx.x] = A[index];
        __syncthreads(); // Synchronize to ensure all data is loaded

        // Calculate tan using shared memory to reduce global memory accesses
        C[index] = tan(sharedA[threadIdx.x]);
    }
}