#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* Licensed to the Apache Software Foundation (ASF) under one
* or more contributor license agreements.  See the NOTICE file
* distributed with this work for additional information
* regarding copyright ownership.  The ASF licenses this file
* to you under the Apache License, Version 2.0 (the
* "License"); you may not use this file except in compliance
* with the License.  You may obtain a copy of the License at
*
*   http://www.apache.org/licenses/LICENSE-2.0
*
* Unless required by applicable law or agreed to in writing,
* software distributed under the License is distributed on an
* "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
* KIND, either express or implied.  See the License for the
* specific language governing permissions and limitations
* under the License.
*/

/**********************************
When updating a kernel or adding a new one,
please compile the ptx file and commit it:
nvcc -ptx -arch=sm_30 SystemML.cu
***********************************/


/**
* Does a copy of upper to lower triangle of the given matrix
* @param ret the input and output array allocated on the GPU
* @param dim the number of rows of the square matrix ret
* @param N total number of elements of the matrix
*/
extern "C"

__global__ void bias_add(double* input, double* bias, double* ret, int rlen, int clen, int PQ) {
    // Calculate the row and column indices
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check for boundaries
    if(ix < rlen && iy < clen) {
        int index = ix * clen + iy;        // Compute the flattened index for the input/output arrays
        int biasIndex = iy / PQ;           // Compute the bias index based on PQ
        ret[index] = input[index] + bias[biasIndex];  // Add the bias to the input and store in the output
    }
}