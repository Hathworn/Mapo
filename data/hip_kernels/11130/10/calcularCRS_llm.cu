#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcularCRS(int *val, int *col_ind, int *row_ptr, int *u, int *resultado, int l) {
    // Calculate unique global thread index
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if (globalIndex < l) { // Ensure within bounds
        int suma = 0;
        
        // Use shared memory for commonly accessed data to optimize memory access
        __shared__ int shared_u[blockDim.y];
        
        // Load data into shared memory
        if (threadIdx.y < l) {
            shared_u[threadIdx.y] = u[threadIdx.y + ((col_ind[row_ptr[globalIndex] - 1] - 1) * l)];
        }
        
        __syncthreads(); // Synchronize threads within block

        // Sum using shared data
        for (int k = row_ptr[globalIndex] - 1; k < row_ptr[globalIndex + 1] - 1; k++) {
            suma += val[k] * shared_u[col_ind[k] - 1];
        }

        // Store the result
        resultado[globalIndex * l + threadIdx.y] = suma;
    }
}