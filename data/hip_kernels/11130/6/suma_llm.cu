#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void suma(int *a, int *b, int *c, int n, int m) {
    // Use flattened block indexing for consistent thread allocation
    int index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.y * gridDim.x;
    if (index < n * m) {
        c[index] = a[index] + b[index];
    }
}