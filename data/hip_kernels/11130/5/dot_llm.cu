#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(int *a, int *b, int *c) {
    __shared__ int temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Compute the partial dot product for each thread
    temp[threadIdx.x] = a[index] * b[index];
    __syncthreads();

    // Use efficient reduction within each block
    for (unsigned int stride = THREADS_PER_BLOCK / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            temp[threadIdx.x] += temp[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Sum results from all blocks using atomic operation
    if (threadIdx.x == 0) {
        atomicAdd(c, temp[0]);
    }
}