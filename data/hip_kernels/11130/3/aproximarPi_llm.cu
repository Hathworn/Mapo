#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void aproximarPi(float *x, float *y, int *z) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Linearize thread index in x direction
    int idy = blockIdx.y * blockDim.y + threadIdx.y; // Linearize thread index in y direction
    int index = idx + idy * N; // Calculate global index

    // Optimize: Use a local variable to reduce global memory accesses
    if ((x[index] * x[index] + y[index] * y[index]) <= 1.0f) {
        atomicAdd(z, 1);
    }
}