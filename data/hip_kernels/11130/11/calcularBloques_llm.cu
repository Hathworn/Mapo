#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcularBloques(int *matriz, int *u, int *resultado, int num_bloques, int nc, int m ){
    int index1 = threadIdx.x + blockIdx.x * blockDim.x;
    int index2 = threadIdx.y + blockIdx.y * blockDim.y;

    // Optimize using register for suma
    register int suma;

    for (int i = 0; i < num_bloques; i++) {
        suma = 0;
        int base_index = index1 * nc;
        int u_base_index = index2 + m * i * nc;
        for (int l = 0; l < nc; l++) {
            suma += matriz[base_index + l] * u[u_base_index + m * l];
        }
        resultado[u_base_index + m * index1] = suma;
    }
}