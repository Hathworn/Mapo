#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot( int *a, int *b, int *c ) {
    __shared__ int temp[THREADS_PER_BLOCK];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Compute product and store in shared memory
    temp[threadIdx.x] = a[index] * b[index];
    __syncthreads();
    
    // Use warp reduction for faster summation
    int sum = 0;
    for (int offset = blockDim.x / 2; offset > 0; offset /= 2) {
        if (threadIdx.x < offset) {
            temp[threadIdx.x] += temp[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Perform atomic add only by the first thread
    if (threadIdx.x == 0) {
        atomicAdd(c, temp[0]);
    }
}