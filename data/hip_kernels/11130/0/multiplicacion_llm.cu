#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define N 100
#define T 4

void llenarMatriz(int*);

__global__ void multiplicacion(int *a, int *b, int *c) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the row index
    int j = blockIdx.y * blockDim.y + threadIdx.y; // Calculate the column index

    if (i < N && j < N) { // Ensure bounds are within matrix size
        int sum = 0; // Temp variable to store sum
        for (int k = 0; k < N; k++) {
            sum += a[k + i * N] * b[j + k * N]; // Accumulate product
        }
        c[j + i * N] = sum; // Store result in c
    }
}