#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiplicacion(int *a, int *b, int *c, int n, int m, int l) {
    // Calculate the row and column index for the current thread
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    // Boundary check to avoid accessing out-of-bounds memory
    if (i < n && j < l) {
        int temp = 0;  // Initialize temporary accumulator variable
        for (int k = 0; k < m; k++) {
            // Accumulate the product of the corresponding elements
            temp += a[k + i * m] * b[j + k * l];
        }
        c[j + i * l] = temp;  // Store the result in matrix c
    }
}