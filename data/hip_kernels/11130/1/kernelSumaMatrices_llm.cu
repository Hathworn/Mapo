#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function for matrix addition
__global__ void kernelSumaMatrices(float *a, float *b, int m, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    // Unroll the loops to improve memory access patterns
    for(; i < m; i += blockDim.x * gridDim.x) {
        for(int j_start = j; j_start < n; j_start += blockDim.y * gridDim.y) {
            a[i * n + j_start] += b[i * n + j_start];
        }
    }
}