#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void aproximarPi(float *x, float *y, int *z, int tam) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    int index = j + i * tam;

    if (index < tam * tam) { // Ensure index is within bounds
        float x_val = x[index];
        float y_val = y[index];
        
        // Reduce repeated calculations
        if ((x_val * x_val + y_val * y_val) <= 1.0f) {
            atomicAdd(z, 1); // Efficiently use atomicAdd
        }
    }
}