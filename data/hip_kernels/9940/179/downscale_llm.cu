#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv( int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x = ii / (d2 * d3);
    int y = (ii / d3) % d2;
    int z = ii % d3;
    int w = ((ii / (d1 * d2 * d3)) % d1) * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x = ii / (d2 * d3);
    int y = (ii / d3) % d2;
    int z = ii % d3;
    int w = ii / (d1 * d2 * d3);
    w /= scale_factor;
    z /= scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x + threadIdx.y * blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
    if (ii >= no_elements) return;
    // Use shared memory for output accumulation to optimize memory access
    __shared__ float buffer[1024]; // Launch with thread blocks sized properly
    buffer[ii] = 0;
    for (int i = 0; i < scale_factor; ++i) {
        for (int j = 0; j < scale_factor; ++j) {
            int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
            buffer[ii] += gradOutput_data[ipidx];
        }
    }
    gradInput_data[ii] = buffer[ii]; // Write back accumulated result 
}