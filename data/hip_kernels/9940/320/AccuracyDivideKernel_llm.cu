#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function by replacing atomic operation to avoid data race
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    if (threadIdx.x == 0) {  // Use only one thread to perform division
        *accuracy /= N;
    }
}