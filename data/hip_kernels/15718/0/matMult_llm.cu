#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 32
#define N 2048

__global__ void matMult(float* A, float* B, float* C) {
    // Block and thread indices
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE]; // Shared memory for A
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE]; // Shared memory for B

    // Initialize sum
    float sum = 0.0;

    // Compute global indices
    int row = by * BLOCK_SIZE + ty;
    int col = bx * BLOCK_SIZE + tx;

    for (int m = 0; m < N / BLOCK_SIZE; ++m) {
        // Load A and B in shared memory
        As[ty][tx] = A[row * N + m * BLOCK_SIZE + tx];
        Bs[ty][tx] = B[(m * BLOCK_SIZE + ty) * N + col];

        // Synchronize threads
        __syncthreads();

        // Compute partial sum
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            sum += As[ty][k] * Bs[k][tx];
        }

        // Synchronize threads
        __syncthreads();
    }

    // Write result to C
    C[row * N + col] = sum;
}