#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void testMemset(float* array, float value, int N) {
    // Utilize HIP built-in function to optimize thread indexing
    int i = hipThreadIdx_x + hipBlockIdx_x * hipBlockDim_x; 

    if (i < N) {
        // Direct assignment to target memory location
        array[i] = value;
    }
}