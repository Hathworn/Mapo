#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(const unsigned char * src, unsigned char * dst, int level, const size_t width, const size_t height)
{
    const size_t xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    if (xIndex < width && yIndex < height)
    {
        size_t o = yIndex * width + xIndex;
        unsigned char srcVal = *(src + o);
        
        // Use ternary operator to eliminate branch 'if'
        * (dst + o) = (level == 256) ? 0 : ((srcVal >= level) ? 255 : 0); 
        // Alternative bitwise operation to avoid branch
        // * (dst + o) = level == 256 ? 0 : ~((srcVal - level - 1) >> 7);
    }
}