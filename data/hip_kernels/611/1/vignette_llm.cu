#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vignette(const unsigned char * __restrict__ src, unsigned char * __restrict__ dst, float inner, float outer, const size_t width, const size_t height)
{
    // Calculate pixel coordinates using block and thread indices
    size_t xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    size_t yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if indices are within image bounds
    if (xIndex < width && yIndex < height)
    {
        // Compute linear offset for 1D array indexing
        size_t offset = yIndex * width + xIndex;
        // Normalize coordinates with respect to image center
        float x = (float)(xIndex - width / 2) / height;
        float y = (float)(yIndex - height / 2) / height;
        // Calculate distance from center
        float d = sqrtf(x * x + y * y);

        // Apply vignette effect based on distance
        if (d < inner)
        {
            // No change if distance less than inner boundary
            dst[offset] = src[offset];
        }
        else if (d > outer)
        {
            // Set pixel to black if distance greater than outer boundary
            dst[offset] = 0;
        }
        else
        {
            // Calculate vignette factor for distances between inner and outer
            float v = 1.0f - (d - inner) / (outer - inner);
            dst[offset] = (unsigned char)(src[offset] * v);
        }
    }
}