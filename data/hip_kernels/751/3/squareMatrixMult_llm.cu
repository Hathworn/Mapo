#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16

__global__ void squareMatrixMult(float *d_a, float *d_b, float *d_result, int n)
{
    __shared__ float tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    float tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub) {
        // Load tiles efficiently
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        tile_a[threadIdx.y][threadIdx.x] = (idx < n * n) ? d_a[idx] : 0.0f;

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        tile_b[threadIdx.y][threadIdx.x] = (idx < n * n) ? d_b[idx] : 0.0f;

        __syncthreads();

        // Unroll loop to improve performance
        #pragma unroll
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Write back the result if within bounds
    if (row < n && col < n) {
        d_result[row * n + col] = tmp;
    }
}