#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixTriUpper(float *a, int m, int n) {
    // Use block and thread index for parallelism
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure indices are within matrix bounds
    if (row < m && col < n) {
        // Set elements below the main diagonal to zero
        if (row > col) {
            a[row * n + col] = 0;
        }
    }
}