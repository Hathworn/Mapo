#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMult(float *a, float *b, float *c, int m, int n, int k)
{
    __shared__ float tile_a[32][32]; // Allocate shared memory for tiles
    __shared__ float tile_b[32][32];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    for (int tileIdx = 0; tileIdx < (n + 31) / 32; ++tileIdx) {
        // Load data into shared memory with boundary check
        if (row < m && tileIdx * 32 + threadIdx.x < n)
            tile_a[threadIdx.y][threadIdx.x] = a[row * n + tileIdx * 32 + threadIdx.x];
        else
            tile_a[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < k && tileIdx * 32 + threadIdx.y < n)
            tile_b[threadIdx.y][threadIdx.x] = b[(tileIdx * 32 + threadIdx.y) * k + col];
        else
            tile_b[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads(); // Synchronize to ensure all data is loaded

        // Compute partial product for current tile
        for (int i = 0; i < 32; ++i)
            sum += tile_a[threadIdx.y][i] * tile_b[i][threadIdx.x];

        __syncthreads(); // Synchronize before loading new tile
    }

    // Store final result
    if (row < m && col < k)
        c[row * k + col] = sum;
}