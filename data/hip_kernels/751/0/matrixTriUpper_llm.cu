#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixTriUpper(float *a, int m, int n) {
    // Calculate row and column using block and thread indices
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds
    if (i < m && j < n) {
        // Only process lower triangle elements
        if (i > j) {
            a[i * n + j] = 0;
        }
    }
}