#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtract_arrays_kernel(int32_t* __restrict a, const int32_t* __restrict b, const int64_t array_count) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Using a single global index calculation
    if (i < array_count) {
        // Check if index is within bounds and perform operation
        a[i] -= b[i];
    }
}