#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_arrays_kernel(int32_t* __restrict a, const int32_t* __restrict b, const int64_t array_count) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a stride loop for scalability
    int stride = blockDim.x * gridDim.x;

    // Loop through the data with calculated stride
    for (; i < array_count; i += stride) {
        a[i] += b[i];
    }
}