#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtract_arrays_kernel(int64_t* __restrict a, const int64_t* __restrict b, const int64_t array_count) {
    // Use a single stride loop with direct calculation to minimize idle threads
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (; i < array_count; i += stride) {
        a[i] -= b[i];
    }
}