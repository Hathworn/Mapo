#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_arrays_kernel(int64_t* __restrict a, const int64_t* __restrict b, const int64_t array_count) {
    // Calculate global index for the current thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by using a single if condition instead of a loop
    if (index < array_count) {
        a[index] += b[index];
    }
}