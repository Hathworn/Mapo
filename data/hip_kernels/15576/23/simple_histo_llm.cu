#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Calculate global thread ID
    int myId = threadIdx.x + blockDim.x * blockIdx.x;

    // Avoid reading out of bounds
    if (myId < BIN_COUNT) {
        int myItem = d_in[myId];
        int myBin = myItem % BIN_COUNT;
        // Perform atomic addition to ensure thread safety
        atomicAdd(&(d_bins[myBin]), 1);
    }
}