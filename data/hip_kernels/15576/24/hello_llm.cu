#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hello()
{
    int tid = threadIdx.x; // Cache thread index
    printf("Hello world! I'm thread %d\n", tid);
}