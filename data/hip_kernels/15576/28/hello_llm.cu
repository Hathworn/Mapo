#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use __shared__ memory and support multiple blocks
__global__ void hello()
{
    // Get the global thread index
    int globalThreadIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIdx.x == 0) // Ensure only one thread per block prints
    {
        printf("Hello world! I'm a thread in block %d\n", blockIdx.x);
    }

    // Optionally sync threads within the block
    __syncthreads();
}