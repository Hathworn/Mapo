#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use atomic operation to avoid race conditions
__global__ void naive_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int myItem = d_in[myId];
    int myBin = myItem % BIN_COUNT;

    // Atomic addition for thread safety
    atomicAdd(&d_bins[myBin], 1);
}