#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth(float * v_new, const float * v) {
    // Use blockDim and blockIdx for optimized indexing
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;

    // Avoid branching by computing valid neighbor indices
    int myLeftIdx = max(0, myIdx - 1);
    int myRightIdx = min(numThreads - 1, myIdx + 1);

    // Load elements from global memory once
    float myElt = v[myIdx];
    float myLeftElt = v[myLeftIdx];
    float myRightElt = v[myRightIdx];

    // Store result in global memory
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
}