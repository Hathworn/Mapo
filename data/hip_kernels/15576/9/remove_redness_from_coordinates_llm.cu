#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void remove_redness_from_coordinates(const unsigned int* d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, 
    int num_coordinates, int num_pixels_y, int num_pixels_x, int template_half_height, int template_half_width) 
{
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;
   
    if (global_index_1d < num_coordinates) 
    {
        // Precompute repetitive variables for performance
        int imgSize = num_pixels_x * num_pixels_y;
        unsigned int image_index_1d = d_coordinates[imgSize - global_index_1d - 1];
        ushort2 image_index_2d = make_ushort2(image_index_1d % num_pixels_x, image_index_1d / num_pixels_x);

        for (int y = image_index_2d.y - template_half_height; y <= image_index_2d.y + template_half_height; y++) 
        {
            for (int x = image_index_2d.x - template_half_width; x <= image_index_2d.x + template_half_width; x++) 
            {
                // Use min/max functions on coordinates to ensure they stay within bounds
                int x_clamped = min(num_pixels_x - 1, max(0, x));
                int y_clamped = min(num_pixels_y - 1, max(0, y));

                // Compute the clamped linear index once for performance
                int image_offset_index_1d_clamped = (num_pixels_x * y_clamped) + x_clamped;

                unsigned char g_value = d_g[image_offset_index_1d_clamped];
                unsigned char b_value = d_b[image_offset_index_1d_clamped];

                // Compute gb average directly without explicit variable
                d_r_output[image_offset_index_1d_clamped] = (g_value + b_value) / 2;
            }
        }
    }
}