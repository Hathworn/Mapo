#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tonemap(float* __restrict__ d_x, float* __restrict__ d_y, float* __restrict__ d_log_Y, float* __restrict__ d_cdf_norm, float* __restrict__ d_r_new, float* __restrict__ d_g_new, float* __restrict__ d_b_new, float min_log_Y, float max_log_Y, float log_Y_range, int num_bins, int num_pixels_y, int num_pixels_x)
{
    int nx = num_pixels_x;
    int ny = num_pixels_y;
    int2 image_index_2d = make_int2(blockIdx.x * blockDim.x + threadIdx.x, blockIdx.y * blockDim.y + threadIdx.y);
    int image_index_1d = nx * image_index_2d.y + image_index_2d.x;

    // Improved boundary check
    if (image_index_2d.x >= nx || image_index_2d.y >= ny) return;

    // Use shared memory for frequent accesses
    __shared__ float shared_cdf_norm[1024];  // Assuming blockDim.x * blockDim.y <= 1024
    if (threadIdx.x < num_bins) {
        shared_cdf_norm[threadIdx.x] = d_cdf_norm[threadIdx.x];
    }
    __syncthreads();

    float x = d_x[image_index_1d];
    float y = d_y[image_index_1d];
    float log_Y = d_log_Y[image_index_1d];
    int bin_index = min(num_bins - 1, int(num_bins * (log_Y - min_log_Y) / log_Y_range));
    float Y_new = shared_cdf_norm[bin_index];

    // Pre-compute reciprocal for dividing multiple times
    float recip_y = 1.0f / y;
    float X_new = x * (Y_new * recip_y);
    float Z_new = (1.0f - x - y) * (Y_new * recip_y);

    // Constants for color transformation
    const float c0 = 3.2406f, c1 = -1.5372f, c2 = -0.4986f;
    const float c3 = -0.9689f, c4 = 1.8758f, c5 = 0.0415f;
    const float c6 = 0.0557f, c7 = -0.2040f, c8 = 1.0570f;

    float r_new = X_new * c0 + Y_new * c1 + Z_new * c2;
    float g_new = X_new * c3 + Y_new * c4 + Z_new * c5;
    float b_new = X_new * c6 + Y_new * c7 + Z_new * c8;

    d_r_new[image_index_1d] = r_new;
    d_g_new[image_index_1d] = g_new;
    d_b_new[image_index_1d] = b_new;
}