#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
    // Each thread puts its partial sum into shared memory
    s[threadIdx.x] = p;
    __syncthreads();

    // Reduce within warp
    if (threadIdx.x < 16) {
        s[threadIdx.x] += s[threadIdx.x + 16];
        s[threadIdx.x] += s[threadIdx.x + 8];
        s[threadIdx.x] += s[threadIdx.x + 4];
        s[threadIdx.x] += s[threadIdx.x + 2];
        s[threadIdx.x] += s[threadIdx.x + 1];
    }
    __syncthreads();

    // Return the result from the first thread
    return s[0];
}

__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    unsigned int p = d_in[t];

    // Each thread calculates its partial sum and stores it in shared memory
    unsigned int sr = shared_reduce(p, s);

    // The first thread writes the result to the output
    if (t == 0) {
        *d_out_shared = sr;
    }
}