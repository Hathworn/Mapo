#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// http://en.wikipedia.org/wiki/Bitonic_sort

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    // Optimize sorting logic
    for (int stage = 0; stage <= 5; stage++)
    {
        int stageStep = 1 << stage;
        for (int substage = stage; substage >= 0; substage--)
        {
            int step = 1 << substage;
            int idx = 2 * tid - (tid & (step - 1));
            
            // Conditional swapping
            if ((sdata[idx] > sdata[idx + step]) == (tid & stageStep))
            {
                float temp = sdata[idx];
                sdata[idx] = sdata[idx + step];
                sdata[idx + step] = temp;
            }
        }
        __syncthreads();
    }

    d_out[tid] = sdata[tid];
}