#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shmem_reduce_kernel(float * d_out, const float * d_in)
{
    // Shared memory for the reduction
    extern __shared__ float sdata[];

    unsigned int myId = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int tid = threadIdx.x;

    // Load shared mem from global mem
    if (myId < gridDim.x * blockDim.x) { // Avoid out-of-bounds access
        sdata[tid] = d_in[myId];
    } else {
        sdata[tid] = 0; // Pad with zero to handle non-power-of-two blocks
    }
    __syncthreads(); // Ensure all threads have loaded data

    // Do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Ensure all adds at one stage are done
    }

    // Only thread 0 writes result for this block back to global mem
    if (tid == 0) {
        d_out[blockIdx.x] = sdata[0];
    }
}