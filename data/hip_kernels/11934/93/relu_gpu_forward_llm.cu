#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop unrolling to process multiple elements per thread
    while (tid < N) {
        out[tid] = in[tid] > 0 ? in[tid] : 0;
        tid += blockDim.x * gridDim.x;
    }
}