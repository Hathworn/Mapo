#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary condition first for early exit
    if (tid >= N) return;

    // Use ternary operator for compactness
    ingrad[tid] = (indata[tid] > 0) ? outgrad[tid] : 0.0f;
}