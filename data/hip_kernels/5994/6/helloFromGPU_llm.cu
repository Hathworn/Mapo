#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU(void) {
    // Use threadIdx and blockIdx to create more unique outputs for each thread
    // Reduce contention on printf usage for higher occupancy
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    printf("Hello World from GPU! Thread: %d\n", idx);
}