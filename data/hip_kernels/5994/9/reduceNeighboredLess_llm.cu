#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighboredLess(int *g_idata, int *g_odata, unsigned int n) {
    // Thread id
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Data pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;
    // Thread id out of range
    if (threadIdx.x >= n) return;

    // Ensure the loop doesn't iterate unnecessarily
    for (int stride = 1; stride < blockDim.x / 2; stride *= 2) {
        // First data index of this thread
        int index = 2 * idx * stride;
        // Data add
        if (index + stride < blockDim.x) {
            idata[index] += idata[index + stride];
        }
        __syncthreads();  // Ensure all operations are complete before moving to the next stride
    }

    // Store the result in the output array
    if (threadIdx.x == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}