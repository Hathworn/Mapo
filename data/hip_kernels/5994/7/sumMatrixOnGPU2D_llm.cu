#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, int Nx, int Ny) {
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate single linear index for improved memory access pattern
    unsigned int idx = iy * Nx + ix; 

    if (ix < Nx && iy < Ny) {
        C[idx] = A[idx] + B[idx];
    }
}