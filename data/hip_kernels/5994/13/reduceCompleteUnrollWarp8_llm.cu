#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceCompleteUnrollWarp8(int *g_idata, int *g_odata, unsigned int n) {
    // Thread and data index calculation for 8-element unroll
    int idx = threadIdx.x + blockIdx.x * blockDim.x * 8;
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // Unroll 8 elements per thread
    int sum = 0;
    if (idx + 7 * blockDim.x < n) {
        sum += g_idata[idx];
        sum += g_idata[idx + blockDim.x];
        sum += g_idata[idx + 2 * blockDim.x];
        sum += g_idata[idx + 3 * blockDim.x];
        sum += g_idata[idx + 4 * blockDim.x];
        sum += g_idata[idx + 5 * blockDim.x];
        sum += g_idata[idx + 6 * blockDim.x];
        sum += g_idata[idx + 7 * blockDim.x];
    }
    idata[threadIdx.x] = sum;

    __syncthreads();

    // Reduction in shared memory using a single loop
    for (unsigned int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (threadIdx.x < stride) {
            idata[threadIdx.x] += idata[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Unroll last warp
    if (threadIdx.x < 32) {
        volatile int *vmem = idata;
        vmem[threadIdx.x] += vmem[threadIdx.x + 32];
        vmem[threadIdx.x] += vmem[threadIdx.x + 16];
        vmem[threadIdx.x] += vmem[threadIdx.x + 8];
        vmem[threadIdx.x] += vmem[threadIdx.x + 4];
        vmem[threadIdx.x] += vmem[threadIdx.x + 2];
        vmem[threadIdx.x] += vmem[threadIdx.x + 1];
    }

    // Write the result of this block to global memory
    if (threadIdx.x == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}