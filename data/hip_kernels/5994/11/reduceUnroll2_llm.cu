#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnroll2(int *g_idata, int *g_odata, unsigned int n) {
    // Combined index for both operations in one step
    int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    // Use a shared memory buffer for intra-block reduction
    extern __shared__ int smem[];

    // Load data into shared memory, ensure no out of bounds access
    if (idx < n) smem[threadIdx.x] = g_idata[idx] + (idx + blockDim.x < n ? g_idata[idx + blockDim.x] : 0);
    else smem[threadIdx.x] = 0;  // Handle situations where idx might be out of bounds

    __syncthreads();

    // Perform reduction using shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            smem[threadIdx.x] += smem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        g_odata[blockIdx.x] = smem[0];
    }
}