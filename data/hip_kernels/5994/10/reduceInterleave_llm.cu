#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceInterleave(int *g_idata, int *g_odata, unsigned int n) {
    // Thread id
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Data pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Thread id out of range
    if (idx >= n) return;

    int tid = threadIdx.x; // Cache thread index

    // Present as reduction in shared memory
    extern __shared__ int sdata[];
    sdata[tid] = idata[tid];
    __syncthreads();

    // Loop unrolling for stride 2 optimization
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}