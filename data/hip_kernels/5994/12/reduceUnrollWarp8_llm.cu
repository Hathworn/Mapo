#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrollWarp8(int *g_idata, int *g_odata, unsigned int n) {
    // Get thread id
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Data pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;
    
    // Unrolling blocks
    if (idx + 7 * blockDim.x < n) {
        int el0 = g_idata[idx];
        int el1 = g_idata[idx + blockDim.x];
        int el2 = g_idata[idx + 2 * blockDim.x];
        int el3 = g_idata[idx + 3 * blockDim.x];
        int el4 = g_idata[idx + 4 * blockDim.x];
        int el5 = g_idata[idx + 5 * blockDim.x];
        int el6 = g_idata[idx + 6 * blockDim.x];
        int el7 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = el0 + el1 + el2 + el3 + el4 + el5 + el6 + el7;
    }
    
    __syncthreads();
    
    // Thread id out of range
    if (idx >= n) return;

    // Use shared memory for intermediate results
    __shared__ int smem[256];
    smem[threadIdx.x] = idata[threadIdx.x];
    __syncthreads();

    // Optimize reduction using single loop
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            smem[threadIdx.x] += smem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Only thread 0 writes the result for the block
    if (threadIdx.x == 0) {
        g_odata[blockIdx.x] = smem[0];
    }
}