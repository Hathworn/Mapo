#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n) {
    // Shared memory for block-level synchronization
    extern __shared__ int sdata[];

    // Compute thread id
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;
    
    // Load input into shared memory
    if (idx < n) sdata[tid] = g_idata[idx];
    else sdata[tid] = 0;
    __syncthreads();
    
    // In-place reduction in shared memory
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int index = 2 * stride * tid;
        if (index < blockDim.x) {
            sdata[index] += sdata[index + stride];
        }
        __syncthreads();
    }
    
    // Write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}