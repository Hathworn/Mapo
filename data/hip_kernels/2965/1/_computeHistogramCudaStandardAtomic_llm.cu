#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

using HistType = uint32_t;

enum class Mode {
    CPU,
    OMP,
    OMP_NOATOMIC,
    CUDA,
    CUDA_NOATOMIC,
    CUDA_SHARED,
};

enum class AtomicTypeCuda {
    NONE,
    STANDARD,
    SHARED,
};

__global__ void _computeHistogramCudaStandardAtomic(const uint8_t *__restrict__ bytes, size_t length, HistType *__restrict__ histogram) {
    // Calculate unique index for each thread
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;

    // Optimize by reducing the total number of atomic operations
    // each thread accumulates its results in a local accumulator first.
    HistType localHist[256] = {0};

    for (size_t i = idx; i < length; i += stride) {
        atomicAdd(&(localHist[bytes[i]]), 1u);
    }

    // Update global histogram with accumulated local results, avoiding overlap
    __shared__ HistType sharedHist[256];
    if (threadIdx.x < 256) {
        sharedHist[threadIdx.x] = 0;
    }
    __syncthreads();

    for (int i = threadIdx.x; i < 256; i += blockDim.x) {
        atomicAdd(&sharedHist[i], localHist[i]);
    }
    __syncthreads();

    if (threadIdx.x < 256) {
        atomicAdd(&(histogram[threadIdx.x]), sharedHist[threadIdx.x]);
    }
}