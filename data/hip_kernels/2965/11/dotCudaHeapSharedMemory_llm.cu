#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dotCudaHeapSharedMemory(const float* a, const float* b, float* dest, const size_t length) {
    // Compute unique thread ID
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Allocate shared memory
    extern __shared__ float cache[];
    
    // Each thread computes its partial dot product
    float temp = 0.0f;
    while (tid < length) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;  // Stride over entire grid
    }
    
    // Store partial result in shared memory
    cache[threadIdx.x] = temp;
    
    // Synchronize threads within the block to ensure all have written to shared memory
    __syncthreads();

    // Perform reduction in shared memory
    size_t i = blockDim.x / 2;
    while (i != 0) {
        if (threadIdx.x < i) {
            cache[threadIdx.x] += cache[threadIdx.x + i];
        }
        __syncthreads();
        i /= 2;
    }

    // Store block result in global memory
    if (threadIdx.x == 0) {
        atomicAdd(dest, cache[0]);
    }
}