#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(const int *a, const int *b, int *dest, const size_t length) {
    // Use thread index to increase parallelism
    int tid = blockIdx.x * blockDim.x + threadIdx.x; 
    
    if (tid < length) {
        dest[tid] = a[tid] - b[tid];
    }
}