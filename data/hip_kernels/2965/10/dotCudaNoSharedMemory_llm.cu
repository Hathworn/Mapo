#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dotCudaNoSharedMemory(const float* a, const float* b, float* dest, const size_t length) {
    // Calculate the unique thread index
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize a partial sum variable for each thread
    float partialSum = 0.0f;
    
    // Use a loop to accumulate products into the partial sum
    for (size_t i = idx; i < length; i += blockDim.x * gridDim.x) {
        partialSum += a[i] * b[i];
    }
    
    // Use atomicAdd to safely accumulate partial sums into global memory
    atomicAdd(dest, partialSum);
}