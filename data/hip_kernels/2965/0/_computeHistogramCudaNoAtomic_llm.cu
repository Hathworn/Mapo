#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

using HistType = uint32_t;

enum class Mode {
    CPU,
    OMP,
    OMP_NOATOMIC,
    CUDA,
    CUDA_NOATOMIC,
    CUDA_SHARED,
};

enum class AtomicTypeCuda {
    NONE,
    STANDARD,
    SHARED,
};

// Optimized kernel function for computing histogram using HIP
__global__ void _computeHistogramCudaNoAtomic(const uint8_t *__restrict__ bytes, size_t length, HistType *__restrict__ histogram) {
    // Get global thread index
    size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    // Calculate stride
    size_t stride = blockDim.x * gridDim.x;

    // Use shared memory to minimize global memory access
    __shared__ HistType shared_histogram[256];
    if (threadIdx.x < 256) {
        shared_histogram[threadIdx.x] = 0;
    }
    __syncthreads();

    // Accumulate histogram in shared memory
    for (size_t i = index; i < length; i += stride) {
        atomicAdd(&shared_histogram[bytes[i]], 1);
    }
    __syncthreads();

    // Write back shared memory histogram to global memory
    if (threadIdx.x < 256) {
        atomicAdd(&histogram[threadIdx.x], shared_histogram[threadIdx.x]);
    }
}