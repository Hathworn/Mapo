#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function with shared memory and reduced memory access
__global__ void add(const int *a, const int *b, int *dest, const size_t length) {
    extern __shared__ int shared_a[], shared_b[];

    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    size_t stride = blockDim.x * gridDim.x;

    // Load data into shared memory
    if (tid < length) {
        shared_a[threadIdx.x] = a[tid];
        shared_b[threadIdx.x] = b[tid];
    }
    __syncthreads();

    // Perform computation using shared memory
    for (; tid < length; tid += stride) {
        dest[tid] = shared_a[threadIdx.x] + shared_b[threadIdx.x];
    }
}