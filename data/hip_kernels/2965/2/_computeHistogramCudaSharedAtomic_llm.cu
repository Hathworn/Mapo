#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

using HistType = uint32_t;

enum class Mode {
    CPU,
    OMP,
    OMP_NOATOMIC,
    CUDA,
    CUDA_NOATOMIC,
    CUDA_SHARED,
};

enum class AtomicTypeCuda {
    NONE,
    STANDARD,
    SHARED,
};

__global__ void _computeHistogramCudaSharedAtomic(const uint8_t *__restrict__ bytes, size_t length, HistType *__restrict__ histogram) {
    __shared__ HistType temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads(); // Initialize shared memory

    size_t stride = blockDim.x * gridDim.x;
    size_t idx = threadIdx.x + blockIdx.x * blockDim.x;

    for (; idx < length; idx += stride) {
        atomicAdd(&temp[bytes[idx]], 1u);
        // Accumulate local histogram
    }
    __syncthreads();

    atomicAdd(&histogram[threadIdx.x], temp[threadIdx.x]);
    // Merge block histograms into global histogram
}