#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtract_kernel(float * in1, float * in2, float * out)
{
    // Precompute linear thread id
    unsigned int id = (blockIdx.z * blockDim.z + threadIdx.z) * c_Size.y * c_Size.x +
                      (blockIdx.y * blockDim.y + threadIdx.y) * c_Size.x +
                      (blockIdx.x * blockDim.x + threadIdx.x);

    if (id >= c_Size.x * c_Size.y * c_Size.z)
        return;

    // Perform subtraction
    out[id] = in1[id] - in2[id];
}