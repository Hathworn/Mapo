#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gradient_and_subtract_kernel(float * in, float * grad_x, float * grad_y, float * grad_z)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Exit directly if the current thread is out of bounds
    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
        return;

    // Precompute strides to optimize index calculations
    long int stride_y = c_Size.x; // Number of elements across x
    long int stride_z = c_Size.y * stride_y; // Total number of elements in xy plane

    long int id = k * stride_z + j * stride_y + i;
    long int id_x = id + 1; // Reuse id for x index calculation
    long int id_y = id + stride_y; // Compute y index using precomputed stride
    long int id_z = id + stride_z; // Compute z index using precomputed stride

    // Optimize gradient computations by checking boundaries first
    if (i != (c_Size.x - 1))
        grad_x[id] -= ((in[id_x] - in[id]) / c_Spacing.x);
    if (j != (c_Size.y - 1))
        grad_y[id] -= ((in[id_y] - in[id]) / c_Spacing.y);
    if (k != (c_Size.z - 1))
        grad_z[id] -= ((in[id_z] - in[id]) / c_Spacing.z);
}