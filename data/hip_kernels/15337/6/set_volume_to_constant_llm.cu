#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_volume_to_constant(float *out, float value)
{
    // Calculate unique thread index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idz = blockIdx.z * blockDim.z + threadIdx.z;

    // Load shared memory to reduce global memory accesses
    __shared__ bool shouldProcess[1];
    if (threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0) {
        shouldProcess[0] = (idx < c_Size.x && idy < c_Size.y && idz < c_Size.z);
    }
    __syncthreads();

    // Only access global memory if within bounds
    if (shouldProcess[0]) {
        long int id = (idz * c_Size.y + idy) * c_Size.x + idx;
        out[id] = value;
    }
}