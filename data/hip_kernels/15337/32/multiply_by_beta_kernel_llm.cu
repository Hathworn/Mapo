#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiply_by_beta_kernel(float * input, float * output, float beta)
{
    // Calculate a single linear index for the 3D grid
    unsigned int idx = blockIdx.z * (gridDim.y * gridDim.x * blockDim.x * blockDim.y) +
                       blockIdx.y * (gridDim.x * blockDim.x * blockDim.y) +
                       blockIdx.x * (blockDim.x * blockDim.y) +
                       threadIdx.y * blockDim.x + threadIdx.x;
    
    // Compute stride as the total number of threads
    unsigned int stride = blockDim.x * blockDim.y * blockDim.z * gridDim.x * gridDim.y * gridDim.z;

    // Loop over all necessary elements
    for (unsigned int id = idx; id < c_Size.x * c_Size.y * c_Size.z; id += stride) {
        output[id] = input[id] * beta; // Perform the multiplication
    }
}