#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_normalize_and_add_to_output(float * dev_vol_in, float * dev_vol_out, float * dev_accumulate_weights, float * dev_accumulate_values)
{
    // Use blockIdx * blockDim + threadIdx calculation
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Use return for early exit if out of bounds
    if (i >= c_volSize.x || j >= c_volSize.y || k >= c_volSize.z) return;

    // Optimize index calculation
    long int out_idx = i + (j + k * c_volSize.y) * c_volSize.x;

    float eps = 1e-6;

    // Combine conditions and reduce branches
    float weight = dev_accumulate_weights[out_idx];
    if (c_normalize && abs(weight) > eps)
    {
        dev_vol_out[out_idx] = dev_vol_in[out_idx] + (dev_accumulate_values[out_idx] / weight);
    }
    else
    {
        dev_vol_out[out_idx] = dev_vol_in[out_idx] + (c_normalize ? 0.0f : dev_accumulate_values[out_idx]);
    }
}