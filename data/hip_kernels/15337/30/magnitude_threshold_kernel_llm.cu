#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void magnitude_threshold_kernel(float * grad_x, float * grad_y, float * grad_z, float gamma)
{
    // Calculate global thread ID
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Check if within bounds
    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
        return;

    // Calculate linear index
    long int id = (k * c_Size.y + j) * c_Size.x + i;

    // Calculate square of the gradient magnitude
    float norm_sq = grad_x[id] * grad_x[id] + grad_y[id] * grad_y[id] + grad_z[id] * grad_z[id];

    // Compare squared magnitude against squared gamma to avoid sqrt
    float gamma_sq = gamma * gamma;
    if (norm_sq > gamma_sq)
    {
        // Calculate ratio safely without sqrt
        float ratio = gamma / sqrt(norm_sq);
        grad_x[id] *= ratio;
        grad_y[id] *= ratio;
        grad_z[id] *= ratio;
    }
}