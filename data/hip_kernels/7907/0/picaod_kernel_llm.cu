#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void picaod_kernel(unsigned int *dev_v, long size, unsigned int *temp)
{
    // Calculate global thread index
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Calculate linear offset 
    int offset = x + y * blockDim.x * gridDim.x;

    // Ensure the index is within the range
    if (offset < size) {
        // Use atomic operation to avoid race conditions
        atomicAdd(&(temp[0]), dev_v[offset]);
    }
}