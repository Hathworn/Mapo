#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* __restrict__ row_sum, int32_t* __restrict__ output, int32_t N) {
    // Use shared memory to reduce redundant global memory access
    __shared__ int32_t shared_row_sum;

    // Load the row_sum into shared memory once per block
    if (threadIdx.x == 0) {
        shared_row_sum = -row_sum[blockIdx.x];
    }
    
    // Synchronize threads to ensure shared memory is loaded
    __syncthreads();

    // Compute offset using shared memory data
    for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
        output[blockIdx.x * N + i] = shared_row_sum;
    }
}