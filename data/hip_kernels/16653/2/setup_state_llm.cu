#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize by launching a grid of 1D block
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate unique ID for each thread
    hiprand_init(seed, id, 0, &state[id]); // Initialize state for each thread
}