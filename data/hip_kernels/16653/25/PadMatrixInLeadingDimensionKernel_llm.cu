#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel with coalesced memory access
__global__ void PadMatrixInLeadingDimensionKernel(const int8_t* __restrict__ src, int8_t* __restrict__ dst, int col_src, int col_dst) {
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = global_idx; i < col_src; i += stride) {
        dst[blockIdx.x * col_dst + i] = src[blockIdx.x * col_src + i];
    }
}