#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixAB(const int32_t* __restrict__ row_sum, const int32_t* __restrict__ col_sum, int32_t* __restrict__ output, int32_t K_A_B, int32_t N) {
    int32_t idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        // Precalculate row_offset to avoid redundant computation
        int32_t row_offset = K_A_B - row_sum[blockIdx.x];
        output[blockIdx.x * N + idx] = row_offset - col_sum[idx];
    }
}