#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Use shared memory for better performance
    extern __shared__ int32_t shared_col_sum[];

    // Calculate global index
    int32_t globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load into shared memory
    if (threadIdx.x < N) {
        shared_col_sum[threadIdx.x] = col_sum[threadIdx.x];
    }
    __syncthreads();

    // Perform computation
    for (int32_t i = globalIdx; i < N; i += blockDim.x * gridDim.x) {
        output[blockIdx.x * N + i] = -shared_col_sum[i % N];
    }
}