#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index
    if (idx < num_elements) { // Ensure the thread is within bounds
        int base_index = idx * 3; // Compute base index for output
        to_normalize[base_index] = batch_index;
        to_normalize[base_index + 1] = class_index;
        to_normalize[base_index + 2] = static_cast<int64_t>(original[idx]);
    }
}