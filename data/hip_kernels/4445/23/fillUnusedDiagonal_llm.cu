#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fillUnusedDiagonal(float* Rmatrices, uint32_t numSamples, uint32_t subArraySize, const uint32_t* subArraySizes)
{
    // Calculate global thread index
    int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int bIdx = blockIdx.y * gridDim.x + blockIdx.x;

    if (bIdx < numSamples)
    {
        int subArraySizeLocal = subArraySizes[bIdx];
        int numelRfull = subArraySize * subArraySize;

        if (subArraySize > subArraySizeLocal)
        {
            float* R = &Rmatrices[bIdx * numelRfull];
            float diagEntry = R[subArraySize * subArraySize - 1];

            for (int diagIdx = subArraySizeLocal + tIdx; diagIdx < subArraySize; diagIdx += blockDim.x * gridDim.x)
            {
                // Calculate the index for a diagonal element
                int matrixIdx = diagIdx * (subArraySize + 1);

                // Set diagonal entry
                R[matrixIdx] = diagEntry;
            }
        }
    }
}