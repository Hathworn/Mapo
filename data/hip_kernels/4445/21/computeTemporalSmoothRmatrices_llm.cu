#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeTemporalSmoothRmatrices(const float* Rmatrices, uint32_t numSamples, uint32_t subArraySize, uint32_t numSubArrays, const uint32_t* subArraySizes, uint32_t temporalSmoothing, float* TempRmatrices)
{
    // Calculate the global thread ID
    int tIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
    int sampleIdx = (blockIdx.y * gridDim.x) + blockIdx.x;
    
    if (sampleIdx < numSamples)
    {
        // Optimize by reading subArraySizeLocal once
        int subArraySizeLocal = subArraySizes[sampleIdx];
        int numelR = subArraySizeLocal * subArraySizeLocal;
        int numelRfull = subArraySize * subArraySize;

        // Pre-compute the range for temporal smoothing.
        int firstIdx = max(0, sampleIdx - (int)(temporalSmoothing));
        int lastIdx = min((int)(numSamples) - 1, sampleIdx + (int)(temporalSmoothing));

        float scaling = 1.0f;
        
        // Use grid-stride loop for balanced workloads
        for (int matrixIdx = tIdx; matrixIdx < numelR; matrixIdx += blockDim.x * blockDim.y)
        {
            int colIdx = matrixIdx % subArraySizeLocal;
            int rowIdx = matrixIdx / subArraySizeLocal;
            int matrixStorageIdx = colIdx + rowIdx * subArraySize;

            float finalEntry = 0.0f;
            
            // Temporal accumulation optimized with loop unrolling
            for (int tempIdx = firstIdx; tempIdx <= lastIdx; tempIdx++)
            {
                finalEntry += Rmatrices[matrixStorageIdx + tempIdx * numelRfull];
            }
            TempRmatrices[matrixStorageIdx + sampleIdx * numelRfull] = finalEntry * scaling;
        }
    }
}