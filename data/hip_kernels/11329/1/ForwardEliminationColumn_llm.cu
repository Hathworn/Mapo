#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define uint unsigned int

#ifdef __INTELLISENSE__
void __syncthreads();
#endif // __INTELLISENSE__

void InitCPUData(double** matrices, int size);
void FillHostMatrix(double** matrices, int size);
hipError_t InitGPUData(double** matrices, int **dSize, int size, int **dStride, int stride);
hipError_t CudaMemcpyMatrix(double** matrices, int size, hipMemcpyKind flag);

void ForwardElimination(double* matrix, int size);
void BackwardSubstitute(double* matrix, int size);

__global__ void ForwardEliminationColumn(double* matrix, int* size, int* row, int* stride, int* pivotRow)
{
    int _size = *size;
    int _row = *row;
    int _stride = *stride;
    int _pivotRow = *pivotRow;
    int startColumn = (blockIdx.x * blockDim.x + threadIdx.x) * _stride;

    double pivot = (double)matrix[_pivotRow * (_size + 1) + _pivotRow];
    double belowPivot = (double)matrix[_row * (_size + 1) + _pivotRow];
    double ratio = belowPivot / pivot;

    // Optimize memory access by reducing redundant calculations
    int rowOffset = _row * (_size + 1);
    int pivotRowOffset = _pivotRow * (_size + 1);

    for (int i = 0; i < _stride; ++i)
    {
        int colIndex = startColumn + i;
        if (colIndex < (_size + 1))
        {
            // Use precomputed row offset to access matrix elements
            matrix[rowOffset + colIndex] -= ratio * matrix[pivotRowOffset + colIndex];
        }
    }
}