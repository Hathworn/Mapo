#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"


#define uint unsigned int

#ifdef __INTELLISENSE__
void __syncthreads();
#endif // __INTELLISENSE__

void InitCPUData(double** matrices, int size);
void FillHostMatrix(double** matrices, int size);
hipError_t InitGPUData(double** matrices, int **dSize, int size, int **dStride, int stride);
hipError_t CudaMemcpyMatrix(double** matrices, int size, hipMemcpyKind flag);

void ForwardElimination(double* matrix, int size);
void BackwardSubstitute(double* matrix, int size);


__global__ void ForwardEliminationColumn(double* matrix, int* size, int* row, int* stride, int* pivotRow)
{
int _size			= *size;
int _row			= *row;
int _stride			= *stride;
int _pivotRow		= *pivotRow;
int startColumn		= (blockIdx.x * blockDim.x + threadIdx.x) * _stride;

double pivot		= (double)matrix[_pivotRow * (_size + 1) + _pivotRow];
double belowPivot	= (double)matrix[_row * (_size + 1) + _pivotRow];

double ratio		= belowPivot / pivot;

for (int i = 0; i < _stride; ++i)
{
if (startColumn + i < (_size + 1))
{
matrix[_row * (_size + 1) + startColumn + i] -= (ratio * matrix[_pivotRow * (_size + 1) + startColumn + i]);
__syncthreads();
}
}
}