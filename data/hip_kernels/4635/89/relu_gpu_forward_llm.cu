#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Use more descriptive variable names 
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to potentially improve performance
    __shared__ float sharedMem[1024];
    if (globalIndex < N) {
        sharedMem[threadIdx.x] = in[globalIndex];
        __syncthreads();

        // Perform ReLU operation
        out[globalIndex] = sharedMem[threadIdx.x] > 0 ? sharedMem[threadIdx.x] : 0;
    }
}