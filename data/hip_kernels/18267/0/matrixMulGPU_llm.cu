#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrixMulGPU(int *a, int *b, int *c, int N)
{
    // Allocate shared memory for sub-matrices of size blockDim.x x blockDim.y
    __shared__ int shared_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int shared_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int val = 0;

    // Loop over the sub-matrices of A and B required to compute the block sub-matrix
    for (int tileIdx = 0; tileIdx < (N + BLOCK_SIZE - 1) / BLOCK_SIZE; ++tileIdx) {
        // Load elements into shared memory
        if (row < N && (tileIdx * BLOCK_SIZE + threadIdx.y) < N)
            shared_a[threadIdx.x][threadIdx.y] = a[row * N + tileIdx * BLOCK_SIZE + threadIdx.y];
        else
            shared_a[threadIdx.x][threadIdx.y] = 0;

        if ((tileIdx * BLOCK_SIZE + threadIdx.x) < N && col < N)
            shared_b[threadIdx.x][threadIdx.y] = b[(tileIdx * BLOCK_SIZE + threadIdx.x) * N + col];
        else
            shared_b[threadIdx.x][threadIdx.y] = 0;

        __syncthreads();  // Synchronize to ensure all elements are loaded

        // Multiply the two matrices together
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            val += shared_a[threadIdx.x][k] * shared_b[k][threadIdx.y];
        }
        __syncthreads();  // Synchronize to ensure computation is done before proceeding
    }

    // Write the block sub-matrix to global memory
    if (row < N && col < N) {
        c[row * N + col] = val;
    }
}