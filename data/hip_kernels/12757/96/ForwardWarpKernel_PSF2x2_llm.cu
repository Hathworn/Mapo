#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    
    if (i >= h || j >= w) return;
    
    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Precompute scaled coordinates
    float cx = u[flow_row_offset + j] * time_scale + j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + i + 1.0f;

    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    int tx = (int) px;
    int ty = (int) py;
    float value = src[image_row_offset + j];
    float weight;

    // Check bounds once using a combined condition
    #pragma unroll
    for (int k = 0; k < 4; ++k) {
        if (!(tx < 0 || tx >= w || ty < 0 || ty >= h)) {
            switch (k) {
                case 0: weight = dx * dy; break;
                case 1: weight = (1.0f - dx) * dy; tx -= 1; break;
                case 2: weight = (1.0f - dx) * (1.0f - dy); ty -= 1; break;
                case 3: weight = dx * (1.0f - dy); tx += 1; break;
            }
            _atomicAdd(dst + ty * image_stride + tx, value * weight);
            _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
        }
        if (k == 1) tx += 1;  // Adjust tx after second iteration
        if (k == 3) ty += 2;  // Adjust ty after fourth iteration
    }
}