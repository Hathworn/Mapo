#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Use register variables for faster access
        const short2 loc = loc_[ptidx];
        const float scaled_x = loc.x * scale;
        const float scaled_y = loc.y * scale;

        // Write results to global memory
        x[ptidx] = scaled_x;
        y[ptidx] = scaled_y;
    }
}