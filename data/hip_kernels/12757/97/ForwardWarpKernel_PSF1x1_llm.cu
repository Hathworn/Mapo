#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Calculate linear thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (idx >= w || idy >= h) return; // Return early if out of bounds

    int flow_row_offset = idy * flow_stride;
    int image_row_offset = idy * image_stride;

    float u_ = u[flow_row_offset + idx];
    float v_ = v[flow_row_offset + idx];

    float cx = u_ * time_scale + (float)idx + 1.0f;
    float cy = v_ * time_scale + (float)idy + 1.0f;

    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    float value = src[image_row_offset + idx];

    // Check bounds and accumulate
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}