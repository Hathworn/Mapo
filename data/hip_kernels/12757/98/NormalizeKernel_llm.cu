#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for increased read efficiency
    __shared__ float shared_norm[BLOCK_SIZE * BLOCK_SIZE];

    if (i < h && j < w) {
        const int pos = i * s + j;
        
        // Load normalization factor into shared memory
        shared_norm[threadIdx.y * blockDim.x + threadIdx.x] = normalization_factor[pos];
        __syncthreads();

        float scale = shared_norm[threadIdx.y * blockDim.x + threadIdx.x];
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

        image[pos] *= invScale;
    }
}