#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define bufSize 700000

struct timeval startwtime,endwtime;

float *h_a;			// Table at host
float *d_a;			// Table at device
int tsize=0;		// number of rows or columns
size_t size = 0 ;	// size of table( tsize* tsize * sizeof(float*))
float* test;

void print(float *);
void make_table();
void serial();
void check();
void copytables();

__global__ void Kernel1(float *A, int N, int k) {
    // Compute 1D global index for matrix element access
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Boundary check to ensure indices are within matrix bounds
    if (i < N && j < N) {
        // Perform computation only if beneficial to minimize memory writes
        float temp = A[i * N + k] + A[k * N + j];
        if (A[i * N + j] > temp) {
            A[i * N + j] = temp;
        }
    }
}