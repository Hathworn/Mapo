#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define bufSize 700000

struct timeval startwtime, endwtime;

float *h_a;         // Table at host
float *d_a;         // Table at device
int tsize = 0;      // number of rows or columns
size_t size = 0;    // size of table( tsize * tsize * sizeof(float*))
float* test;

void print(float *);
void make_table();
void serial();
void check();
void copytables();

__global__ void Kernel2(float *A, int N, int k) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i >= N || j >= N) return; // Boundary check

    __shared__ float k_k1, k1_k;

    // Load shared memory efficiently by only one thread
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        k_k1 = A[k * N + (k + 1)];
        k1_k = A[(k + 1) * N + k];
    }

    __syncthreads(); // Ensure shared memory load

    float x = A[k * N + j];
    float y = A[i * N + k];
    float xn = A[i * N + (k + 1)];
    float yn = A[(k + 1) * N + j];

    float asked = A[i * N + j];

    // DkF
    asked = min(asked, x + y);

    // D(k+1)
    xn = min(xn, y + k_k1);

    // (k+1)F
    yn = min(yn, x + k1_k);

    // D(k+1)F
    asked = min(asked, xn + yn);

    // Update matrix with minimum distance
    A[i * N + j] = asked;
}