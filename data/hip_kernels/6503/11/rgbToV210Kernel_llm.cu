#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void rgbToV210Kernel(uint16_t *pSrc, uint16_t *pDst, int nSrcWidth, int nHeight) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uint3 rgb;
    uint4 pF;
    int nDstW = nSrcWidth / 18;
    int nDstH = nHeight;

    if (tid < nDstW && tidd < nDstH) {
        int k = tid * 18;
        int j = tidd * nSrcWidth;

        // Precompute offsets
        int indices[6] = {0, 3, 6, 9, 12, 15};

        // Shared memory for RGB values
        __shared__ uint16_t sharedRGB[18];
        if (threadIdx.x < 18) {
            sharedRGB[threadIdx.x] = pSrc[j + k + threadIdx.x];
        }
        __syncthreads();

        uint32_t yuv[6];
        uint32_t v[3], u[3], y[6];

        #pragma unroll
        for (int i = 0; i < 6; i++) {
            int baseIdx = indices[i];
            rgb.x = sharedRGB[baseIdx];
            rgb.y = sharedRGB[baseIdx + 1];
            rgb.z = sharedRGB[baseIdx + 2];

            // Compute YUV values
            yuv[i] = (rgb.x * 299 + rgb.y * 587 + rgb.z * 114) / 1000;
            if (i < 3) {
                u[i] = (512000 - rgb.x * 169 - rgb.y * 332 + rgb.z * 500) / 1000;
                v[i] = (512000 + rgb.x * 500 - rgb.y * 419 - rgb.z * 81) / 1000;
            }
        }
        
        // Pack values
        pF.x = (v[0] << 20) | (yuv[0] << 10) | u[0];
        pF.y = (yuv[2] << 20) | (u[1] << 10) | yuv[1];
        pF.z = (u[2] << 20) | (yuv[3] << 10) | v[1];
        pF.w = (yuv[5] << 20) | (v[2] << 10) | yuv[4];

        // Write to destination
        k = tid * 8;
        j *= 4;
        j /= 9;
        
        pDst[j + k + 0] = static_cast<uint16_t>(pF.x & 0x0000FFFF);
        pDst[j + k + 1] = static_cast<uint16_t>(pF.x >> 16);
        pDst[j + k + 2] = static_cast<uint16_t>(pF.y & 0x0000FFFF);
        pDst[j + k + 3] = static_cast<uint16_t>(pF.y >> 16);
        pDst[j + k + 4] = static_cast<uint16_t>(pF.z & 0x0000FFFF);
        pDst[j + k + 5] = static_cast<uint16_t>(pF.z >> 16);
        pDst[j + k + 6] = static_cast<uint16_t>(pF.w & 0x0000FFFF);
        pDst[j + k + 7] = static_cast<uint16_t>(pF.w >> 16);
    }
}