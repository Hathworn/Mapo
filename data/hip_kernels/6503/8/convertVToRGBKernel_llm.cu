#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void convertVToRGBKernel(const uint16_t *pV210, uint8_t *tt1, int nSrcWidth, int nDstWidth, int nDstHeight, int *lookupTable) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uint16_t tt[6];
    uint4 pF;
    int nDstH = nDstHeight;
    int nDstW = nSrcWidth / 8;

    if (tid < nDstW && tidd < nDstH) {
        int j = tidd * nSrcWidth;
        int k = tid * 8;
        pF.x = (uint32_t)pV210[j + k + 0] + ((uint32_t)pV210[j + k + 1] << 16);
        pF.y = (uint32_t)pV210[j + k + 2] + ((uint32_t)pV210[j + k + 3] << 16);
        pF.z = (uint32_t)pV210[j + k + 4] + ((uint32_t)pV210[j + k + 5] << 16);
        pF.w = (uint32_t)pV210[j + k + 6] + ((uint32_t)pV210[j + k + 7] << 16);

        // Simplify bit manipulations using masks
        uint32_t v[3], y[6], u[3];
        v[0] = (pF.x >> 20) & 0x3FF;
        y[0] = ((pF.x >> 10) & 0xFFC) * 1000;
        u[0] = pF.x & 0x3FF;

        y[1] = pF.y & 0x3FF * 1000;
        u[1] = (pF.y >> 10) & 0x3FF;

        y[2] = ((pF.y >> 20) & 0xFFC) * 1000;
        v[1] = pF.z & 0x3FF;

        y[3] = ((pF.z >> 10) & 0xFFC) * 1000;
        u[2] = (pF.z >> 20) & 0x3FF;

        y[4] = pF.w & 0x3FF * 1000;
        v[2] = (pF.w >> 10) & 0x3FF;

        y[5] = ((pF.w >> 20) & 0xFFC) * 1000;

        k = tid * 18;
        j = (j * 9) / 4;

#pragma unroll
        for (int i = 0; i < 3; ++i) {
            int r = 1407 * v[i] - 720384;
            int g = 716 * v[i] + 345 * u[i] - 543232;
            int b = 1779 * u[i] - 910848;

            tt[0] = (y[i * 2] + r) / 1000;
            tt[1] = (y[i * 2] - g) / 1000;
            tt[2] = (y[i * 2] + b) / 1000;

            tt[3] = (y[i * 2 + 1] + r) / 1000;
            tt[4] = (y[i * 2 + 1] - g) / 1000;
            tt[5] = (y[i * 2 + 1] + b) / 1000;

#pragma unroll
            for (int j = 0; j < 6; ++j) {
                tt1[j + k + i * 6] = lookupTable[tt[j]];
            }
        }
    }
}