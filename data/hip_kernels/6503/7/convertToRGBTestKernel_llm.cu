#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void convertToRGBTestKernel(const uint16_t *pV210, uint8_t *tt, int nSrcWidth, int nDstWidth, int nDstHeight) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;

    if (tid < (nSrcWidth / 8) && tidd < nDstHeight) {
        // Precompute indices
        int j = (tidd * nSrcWidth) * 9 / 4;
        int k = tid * 18;

        uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
        uint4 pF;

        // Coalesced memory access to load data
        int baseIdx = tidd * nSrcWidth + tid * 8;
        pF.x = (uint32_t)pV210[baseIdx + 0] + ((uint32_t)pV210[baseIdx + 1] << 16);
        pF.y = (uint32_t)pV210[baseIdx + 2] + ((uint32_t)pV210[baseIdx + 3] << 16);
        pF.z = (uint32_t)pV210[baseIdx + 4] + ((uint32_t)pV210[baseIdx + 5] << 16);
        pF.w = (uint32_t)pV210[baseIdx + 6] + ((uint32_t)pV210[baseIdx + 7] << 16);

        // Load and compute YUV components
        v0 = (pF.x >> 20) & 0x3FF;
        y0 = ((pF.x >> 10) & 0x3FF) * 1000;
        u0 = pF.x & 0x3FF;
        y2 = ((pF.y >> 20) & 0x3FF) * 1000;
        u1 = (pF.y >> 10) & 0x3FF;
        y1 = (pF.y & 0x3FF) * 1000;
        u2 = (pF.z >> 20) & 0x3FF;
        y3 = ((pF.z >> 10) & 0x3FF) * 1000;
        v1 = pF.z & 0x3FF;
        y5 = ((pF.w >> 20) & 0x3FF) * 1000;
        v2 = (pF.w >> 10) & 0x3FF;
        y4 = (pF.w & 0x3FF) * 1000;

        // Precompute common multipliers for output conversion
        const float scale = 0.249f / 1000.f;

        // Compute and store RGB for each pixel
        for (int i = 0; i < 2; ++i) {
            // Calculate RGB
            int r = 1407 * ((i == 0) ? v0 : v1) - 720384;
            int g = 716 * ((i == 0) ? v0 : v1) + 345 * ((i == 0) ? u0 : u1) - 543232;
            int b = 1779 * ((i == 0) ? u0 : u1) - 910848;

            int yOffsetIndex = (i == 0) ? 0 : 9;
            int yOffset[2] = {0, 3};

            for (int yIdx = 0; yIdx < 2; ++yIdx) {
                int yComp = ((yIdx == 0) ? y0 : y2) + ((i == 1) ? yOffset[yIdx] : 0);
                tt[j + k + yOffsetIndex + yIdx * 3 + 0] = (yComp + r) * scale;
                tt[j + k + yOffsetIndex + yIdx * 3 + 1] = (yComp - g) * scale;
                tt[j + k + yOffsetIndex + yIdx * 3 + 2] = (yComp + b) * scale;
            }
        }

        // Compute and store RGB for the last two pixels
        int r = 1407 * v2 - 720384;
        int g = 716 * v2 + 345 * u2 - 543232;
        int b = 1779 * u2 - 910848;

        tt[j + k + 12] = (y4 + r) * scale;
        tt[j + k + 13] = (y4 - g) * scale;
        tt[j + k + 14] = (y4 + b) * scale;

        tt[j + k + 15] = (y5 + r) * scale;
        tt[j + k + 16] = (y5 - g) * scale;
        tt[j + k + 17] = (y5 + b) * scale;
    }
}