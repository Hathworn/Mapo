#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void resizeBatchKernel(const uint16_t *p_Src, int nSrcPitch, int nSrcHeight, uint8_t *dpDst0, uint8_t *dpDst1, uint8_t *dpDst2, int nDstWidth, int nDstHeight, int *lookupTable_cuda) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uint4 pF;
    int scale = nSrcHeight / nDstHeight;
    int nDstH = nDstHeight;

    if (scale == 4 || scale == 2 || scale == 6) {
        int nDstW = (scale == 4) ? nDstWidth / 6 : (scale == 6) ? nDstWidth / 2 : nDstWidth / 6;

        // Ensure thread is within the output dimensions
        if (tid < nDstW && tidd < nDstH) {
            int j = tidd * nSrcPitch * scale;
            int k = tid * ((scale == 4) ? 32 : (scale == 6) ? 16 : 16);

            if (scale == 4) {
                // Optimize data gathering by removing redundant memory loads
                pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);
                pF.w = (uint32_t)p_Src[j + k + 6];

                uint32_t v0 = (pF.x >> 20) & 0x3FF;
                uint32_t y0 = (pF.x >> 10) & 0x3FF;
                uint32_t u0 = pF.x & 0x3FF;
                uint32_t y1 = pF.w & 0x3FF;

                pF.y = (uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16);
                pF.z = (uint32_t)p_Src[j + k + 12];

                uint32_t y2 = (pF.y >> 20) & 0x3FF;
                uint32_t u1 = (pF.y >> 10) & 0x3FF;
                uint32_t v1 = pF.z & 0x3FF;

                pF.x = (uint32_t)p_Src[j + k + 16] + ((uint32_t)p_Src[j + k + 17] << 16);
                pF.z = ((uint32_t)p_Src[j + k + 21] << 16);
                pF.w = (uint32_t)p_Src[j + k + 22] + ((uint32_t)p_Src[j + k + 23] << 16);

                uint32_t y3 = (pF.x >> 10) & 0x3FF;
                uint32_t u2 = (pF.z >> 20) & 0x3FF;
                uint32_t v2 = (pF.w >> 10) & 0x3FF;
                uint32_t y4 = pF.w & 0x3FF;

                pF.y = ((uint32_t)p_Src[j + k + 27] << 16);
                uint32_t y5 = (pF.y >> 20) & 0x3FF;

                k = tid * 6;
                j = tidd * nDstWidth;
                dpDst0[j + k + 0] = lookupTable_cuda[y0];
                dpDst0[j + k + 1] = lookupTable_cuda[y1];
                dpDst0[j + k + 2] = lookupTable_cuda[y2];
                dpDst0[j + k + 3] = lookupTable_cuda[y3];
                dpDst0[j + k + 4] = lookupTable_cuda[y4];
                dpDst0[j + k + 5] = lookupTable_cuda[y5];

                k = tid * 3;
                j = tidd * nDstWidth / 2;
                dpDst1[j + k + 0] = lookupTable_cuda[u0];
                dpDst1[j + k + 1] = lookupTable_cuda[u1];
                dpDst1[j + k + 2] = lookupTable_cuda[u2];
                dpDst2[j + k + 0] = lookupTable_cuda[v0];
                dpDst2[j + k + 1] = lookupTable_cuda[v1];
                dpDst2[j + k + 2] = lookupTable_cuda[v2];
            } else if (scale == 6) {
                // Similar optimization for scale == 6
                pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);

                uint32_t v0 = (pF.x >> 20) & 0x3FF;
                uint32_t y0 = (pF.x >> 10) & 0x3FF;
                uint32_t u0 = pF.x & 0x3FF;

                pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);

                uint32_t y1 = (pF.x >> 10) & 0x3FF;

                k = tid * 2;
                j = tidd * nDstWidth;
                dpDst0[j + k + 0] = lookupTable_cuda[y0];
                dpDst0[j + k + 1] = lookupTable_cuda[y1];
                k = tid;
                j = tidd * nDstWidth / 2;
                dpDst1[j + k + 0] = lookupTable_cuda[u0];
                dpDst2[j + k + 1] = lookupTable_cuda[v0];
            } else if (scale == 2) {
                // Optimize data gathering for scale == 2
                pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);
                pF.y = ((uint32_t)p_Src[j + k + 3] << 16);
                pF.z = ((uint32_t)p_Src[j + k + 5] << 16);
                pF.w = (uint32_t)p_Src[j + k + 6] + ((uint32_t)p_Src[j + k + 7] << 16);

                uint32_t v0 = (pF.x >> 20) & 0x3FF;
                uint32_t y0 = (pF.x >> 10) & 0x3FF;
                uint32_t u0 = pF.x & 0x3FF;
                uint32_t y1 = (pF.y >> 20) & 0x3FF;
                uint32_t u1 = (pF.z >> 20) & 0x3FF;
                uint32_t v1 = (pF.w >> 10) & 0x3FF;
                uint32_t y2 = pF.w & 0x3FF;

                pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);
                pF.y = (uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16);
                pF.z = (uint32_t)p_Src[j + k + 12];
                pF.w = (uint32_t)p_Src[j + k + 14];

                uint32_t y3 = (pF.x >> 10) & 0x3FF;
                uint32_t y4 = (pF.y >> 20) & 0x3FF;
                uint32_t u2 = (pF.y >> 10) & 0x3FF;
                uint32_t v2 = pF.z & 0x3FF;
                uint32_t y5 = pF.w & 0x3FF;

                k = tid * 6;
                j = tidd * nDstWidth;
                dpDst0[j + k + 0] = lookupTable_cuda[y0];
                dpDst0[j + k + 1] = lookupTable_cuda[y1];
                dpDst0[j + k + 2] = lookupTable_cuda[y2];
                dpDst0[j + k + 3] = lookupTable_cuda[y3];
                dpDst0[j + k + 4] = lookupTable_cuda[y4];
                dpDst0[j + k + 5] = lookupTable_cuda[y5];

                k = tid * 3;
                j = tidd * nDstWidth / 2;
                dpDst1[j + k + 0] = lookupTable_cuda[u0];
                dpDst1[j + k + 1] = lookupTable_cuda[u1];
                dpDst1[j + k + 2] = lookupTable_cuda[u2];
                dpDst2[j + k + 0] = lookupTable_cuda[v0];
                dpDst2[j + k + 1] = lookupTable_cuda[v1];
                dpDst2[j + k + 2] = lookupTable_cuda[v2];
            }
        }
    }
}