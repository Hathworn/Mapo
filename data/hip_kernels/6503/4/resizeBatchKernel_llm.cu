#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void resizeBatchKernel(const uint16_t *p_Src, int nSrcPitch, int nSrcHeight, uint8_t *dpDst, int nDstWidth, int nDstHeight, int *lookupTable_cuda) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uint4 pF;
    int scale = nSrcHeight / nDstHeight;
    
    const int nDstH = nDstHeight;
    const int nDstW = nDstWidth / ((scale == 6) ? 2 : 6); // Calculate destination width depending on scale
    
    if (tid < nDstW && tidd < nDstH) {
        int j = tidd * nSrcPitch * scale;
        int k = tid * ((scale == 6) ? 16 : 32); // Calculate source offset depending on scale
        
        if (scale == 4) { // Scale is 4, optimized branch
            pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);
            pF.w = (uint32_t)p_Src[j + k + 6];
            
            uint32_t v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
            uint32_t y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            uint32_t u0 = (uint32_t)(pF.x & 0x000003FF);
            uint32_t y1 = (uint32_t)(pF.w & 0x000003FF);
            
            pF.y = (uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16);
            pF.z = (uint32_t)p_Src[j + k + 12];
            
            uint32_t y2 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            uint32_t u1 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
            uint32_t v1 = (uint32_t)(pF.z & 0x000003FF);
            
            pF.x = (uint32_t)p_Src[j + k + 16] + ((uint32_t)p_Src[j + k + 17] << 16);
            pF.z = ((uint32_t)p_Src[j + k + 21] << 16);
            pF.w = (uint32_t)p_Src[j + k + 22] + ((uint32_t)p_Src[j + k + 23] << 16);
            
            uint32_t y3 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            uint32_t u2 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
            uint32_t v2 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
            uint32_t y4 = (uint32_t)(pF.w & 0x000003FF);
            
            pF.y = ((uint32_t)p_Src[j + k + 27] << 16);
            
            uint32_t y5 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            
            k = tid * 6;
            j = tidd * nDstWidth;
            dpDst[j + k + 0] = lookupTable_cuda[y0];
            dpDst[j + k + 1] = lookupTable_cuda[y1];
            dpDst[j + k + 2] = lookupTable_cuda[y2];
            dpDst[j + k + 3] = lookupTable_cuda[y3];
            dpDst[j + k + 4] = lookupTable_cuda[y4];
            dpDst[j + k + 5] = lookupTable_cuda[y5];
            
            k = tid * 3;
            j = tidd * nDstWidth / 2;
            dpDst[j + k + 0] = lookupTable_cuda[u0];
            dpDst[j + k + 1] = lookupTable_cuda[u1];
            dpDst[j + k + 2] = lookupTable_cuda[u2];
            
            j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
            dpDst[j + k + 0] = lookupTable_cuda[v0];
            dpDst[j + k + 1] = lookupTable_cuda[v1];
            dpDst[j + k + 2] = lookupTable_cuda[v2];
        }
        else if (scale == 6) { // Scale is 6, optimized branch
            pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);
            
            uint32_t v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
            uint32_t y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            uint32_t u0 = (uint32_t)(pF.x & 0x000003FF);
            
            pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);
            
            uint32_t y1 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            
            k = tid * 2;
            j = tidd * nDstWidth;
            dpDst[j + k + 0] = lookupTable_cuda[y0];
            dpDst[j + k + 1] = lookupTable_cuda[y1];
            
            k = tid;
            j = tidd * nDstWidth / 2;
            dpDst[j + k + 0] = lookupTable_cuda[u0];
            j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
            dpDst[j + k + 1] = lookupTable_cuda[v0];
        }
        else if (scale == 2) { // Scale is 2, optimized branch
            k = tid * 16;
            
            pF.x = (uint32_t)p_Src[j + k + 0] + ((uint32_t)p_Src[j + k + 1] << 16);
            pF.y = ((uint32_t)p_Src[j + k + 3] << 16);
            pF.z = ((uint32_t)p_Src[j + k + 5] << 16);
            pF.w = (uint32_t)p_Src[j + k + 6] + ((uint32_t)p_Src[j + k + 7] << 16);
            
            uint32_t v0 = (uint32_t)((pF.x & 0x3FF00000) >> 20);
            uint32_t y0 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            uint32_t u0 = (uint32_t)(pF.x & 0x000003FF);
            uint32_t y1 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            uint32_t u1 = (uint32_t)((pF.z & 0x3FF00000) >> 20);
            uint32_t v1 = (uint32_t)((pF.w & 0x000FFC00) >> 10);
            uint32_t y2 = (uint32_t)(pF.w & 0x000003FF);
            
            pF.x = (uint32_t)p_Src[j + k + 8] + ((uint32_t)p_Src[j + k + 9] << 16);
            pF.y = (uint32_t)p_Src[j + k + 10] + ((uint32_t)p_Src[j + k + 11] << 16);
            pF.z = (uint32_t)p_Src[j + k + 12];
            pF.w = (uint32_t)p_Src[j + k + 14];
            
            uint32_t y3 = (uint32_t)((pF.x & 0x000FFC00) >> 10);
            uint32_t y4 = (uint32_t)((pF.y & 0x3FF00000) >> 20);
            uint32_t u2 = (uint32_t)((pF.y & 0x000FFC00) >> 10);
            uint32_t v2 = (uint32_t)(pF.z & 0x000003FF);
            uint32_t y5 = (uint32_t)(pF.w & 0x000003FF);
            
            k = tid * 6;
            j = tidd * nDstWidth;
            dpDst[j + k + 0] = lookupTable_cuda[y0];
            dpDst[j + k + 1] = lookupTable_cuda[y1];
            dpDst[j + k + 2] = lookupTable_cuda[y2];
            dpDst[j + k + 3] = lookupTable_cuda[y3];
            dpDst[j + k + 4] = lookupTable_cuda[y4];
            dpDst[j + k + 5] = lookupTable_cuda[y5];
            
            k = tid * 3;
            j = tidd * nDstWidth / 2;
            dpDst[j + k + 0] = lookupTable_cuda[u0];
            dpDst[j + k + 1] = lookupTable_cuda[u1];
            dpDst[j + k + 2] = lookupTable_cuda[u2];
            
            j = tidd * nDstWidth / 2 + nDstWidth * nDstHeight * 3 / 2;
            dpDst[j + k + 0] = lookupTable_cuda[v0];
            dpDst[j + k + 1] = lookupTable_cuda[v1];
            dpDst[j + k + 2] = lookupTable_cuda[v2];
        }
    }
}