#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void resizeRGBKernel(const uint16_t *p_Src, int nSrcPitch, int nSrcHeight, uint8_t *p_dst, int nDstWidth, int nDstHeight, int *lookupTable_cuda) {
    // Calculate pixel coordinates in the destination image
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds for destination image
    if (tid < nDstWidth && tidd < nDstHeight) {
        // Compute scales
        int yScale = nSrcHeight / nDstHeight;
        int xScale = nSrcPitch / nDstWidth;

        // Compute source image index
        int srcIndex = tidd * yScale * nSrcPitch * 3 + tid * xScale * 3;
        
        // Load RGB values
        ushort3 rgb;
        rgb.x = p_Src[srcIndex + 0];
        rgb.y = p_Src[srcIndex + 1];
        rgb.z = p_Src[srcIndex + 2];
        
        // Compute destination image index
        int dstIndex = tidd * nDstWidth * 3 + tid * 3;
        
        // Apply lookup table and store result
        p_dst[dstIndex + 0] = lookupTable_cuda[rgb.x];
        p_dst[dstIndex + 1] = lookupTable_cuda[rgb.y];
        p_dst[dstIndex + 2] = lookupTable_cuda[rgb.z];
    }
}