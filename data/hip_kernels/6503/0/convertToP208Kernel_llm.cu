#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void convertToP208Kernel(uint16_t *pV210, uint16_t *dP208, int nPitch, int nWidth, int nHeight) {
    // Calculate thread ids
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;

    // Constants and optimizations
    uint32_t pF[4];
    int nDstW = nPitch / 8;
    int nDstH = nHeight;

    // Ensure work is done only within bounds
    if (tid < nDstW && tidd < nDstH) {
        int k = tid * 8;
        int j = tidd * nPitch;
        
        // Optimize memory accesses, coalescing reads of pV210
        pF[0] = __ldg(&pV210[j + k + 0]) + ((uint32_t)__ldg(&pV210[j + k + 1]) << 16);
        pF[1] = __ldg(&pV210[j + k + 2]) + ((uint32_t)__ldg(&pV210[j + k + 3]) << 16);
        pF[2] = __ldg(&pV210[j + k + 4]) + ((uint32_t)__ldg(&pV210[j + k + 5]) << 16);
        pF[3] = __ldg(&pV210[j + k + 6]) + ((uint32_t)__ldg(&pV210[j + k + 7]) << 16);

        // Avoid redundant calculations
        uint32_t v0 = (pF[0] & 0x3FF00000) >> 20;
        uint32_t y0 = (pF[0] & 0x000FFC00) >> 10;
        uint32_t u0 = pF[0] & 0x000003FF;
        uint32_t y2 = (pF[1] & 0x3FF00000) >> 20;
        uint32_t u1 = (pF[1] & 0x000FFC00) >> 10;
        uint32_t y1 = pF[1] & 0x000003FF;
        uint32_t u2 = (pF[2] & 0x3FF00000) >> 20;
        uint32_t y3 = (pF[2] & 0x000FFC00) >> 10;
        uint32_t v1 = pF[2] & 0x000003FF;
        uint32_t y5 = (pF[3] & 0x3FF00000) >> 20;
        uint32_t v2 = (pF[3] & 0x000FFC00) >> 10;
        uint32_t y4 = pF[3] & 0x000003FF;

        // Improved indexing for dP208
        k = tid * 6;
        j = tidd * nPitch * 3 / 4;
        dP208[j + k + 0] = y0;
        dP208[j + k + 1] = y1;
        dP208[j + k + 2] = y2;
        dP208[j + k + 3] = y3;
        dP208[j + k + 4] = y4;
        dP208[j + k + 5] = y5;

        // Optimized loop for U and V values
        k = tid * 3;
        j = tidd * nPitch * 3 / 8 + nWidth * nHeight;
        dP208[j + k + 0] = u0;
        dP208[j + k + 1] = u1;
        dP208[j + k + 2] = u2;
        j = tidd * nPitch * 3 / 8 + nWidth * nHeight * 3 / 2;
        dP208[j + k + 0] = v0;
        dP208[j + k + 1] = v1;
        dP208[j + k + 2] = v2;
    }
}