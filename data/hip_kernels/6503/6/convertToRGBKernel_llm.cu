#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void convertToRGBKernel(const uint16_t *pV210, uint16_t *tt, int nSrcWidth, int nDstWidth, int nDstHeight) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
    uint4 pF;

    if (tid < (nSrcWidth / 8) && tidd < nDstHeight) {
        int j = tidd * nSrcWidth;
        int k = tid * 8;
        
        // Use float2 to load data for better performance
        float2 p0 = *reinterpret_cast<const float2*>(&pV210[j + k]);
        float2 p1 = *reinterpret_cast<const float2*>(&pV210[j + k + 4]);
        
        pF.x = __float2uint_rz(p0.x * 65536.0f);
        pF.y = __float2uint_rz(p0.y * 65536.0f);
        pF.z = __float2uint_rz(p1.x * 65536.0f);
        pF.w = __float2uint_rz(p1.y * 65536.0f);

        v0 = (pF.x >> 20) & 0x3FF;
        y0 = ((pF.x >> 10) & 0x3FF) * 1000;
        u0 = pF.x & 0x3FF;
        y2 = ((pF.y >> 20) & 0x3FF) * 1000;
        u1 = (pF.y >> 10) & 0x3FF;
        y1 = (pF.y & 0x3FF) * 1000;
        u2 = (pF.z >> 20) & 0x3FF;
        y3 = ((pF.z >> 10) & 0x3FF) * 1000;
        v1 = pF.z & 0x3FF;
        y5 = ((pF.w >> 20) & 0x3FF) * 1000;
        v2 = (pF.w >> 10) & 0x3FF;
        y4 = (pF.w & 0x3FF) * 1000;

        k = tid * 18;
        j = (j * 9) / 4;

        // Pre-compute conversion factors
        int r = 1407 * v0 - 720384, g = 716 * v0 + 345 * u0 - 543232, b = 1779 * u0 - 910848;
        int outIdx = j + k;
        
        // Use loop unrolling for better performance
        tt[outIdx] = (y0 + r) / 1000;
        tt[outIdx + 1] = (y0 - g) / 1000;
        tt[outIdx + 2] = (y0 + b) / 1000;
        
        tt[outIdx + 3] = (y1 + r) / 1000;
        tt[outIdx + 4] = (y1 - g) / 1000;
        tt[outIdx + 5] = (y1 + b) / 1000;

        r = 1407 * v1 - 720384;
        g = 716 * v1 + 345 * u1 - 543232;
        b = 1779 * u1 - 910848;
        
        tt[outIdx + 6] = (y2 + r) / 1000;
        tt[outIdx + 7] = (y2 - g) / 1000;
        tt[outIdx + 8] = (y2 + b) / 1000;
        
        tt[outIdx + 9] = (y3 + r) / 1000;
        tt[outIdx + 10] = (y3 - g) / 1000;
        tt[outIdx + 11] = (y3 + b) / 1000;

        r = 1407 * v2 - 720384;
        g = 716 * v2 + 345 * u2 - 543232;
        b = 1779 * u2 - 910848;
        
        tt[outIdx + 12] = (y4 + r) / 1000;
        tt[outIdx + 13] = (y4 - g) / 1000;
        tt[outIdx + 14] = (y4 + b) / 1000;

        tt[outIdx + 15] = (y5 + r) / 1000;
        tt[outIdx + 16] = (y5 - g) / 1000;
        tt[outIdx + 17] = (y5 + b) / 1000;
    }
}