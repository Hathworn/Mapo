#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ static void convertPToRGBKernel(const uint16_t * __restrict__ dpSrc, uint8_t * __restrict__ tt1, int nSrcWidth, int nDstWidth, int nDstHeight, int * __restrict__ lookupTable) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;

    if (tid < nSrcWidth / 2 && tidd < nDstHeight) { // Compute tid limits
        int kSrc = tid * 2;
        int jSrc = tidd * nSrcWidth;

        uint32_t y0 = static_cast<uint32_t>(dpSrc[jSrc + kSrc + 0]) * 1000;
        uint32_t y1 = static_cast<uint32_t>(dpSrc[jSrc + kSrc + 1]) * 1000;
        
        int kUV = tid;
        int jUSrc = tidd * nSrcWidth / 2 + nDstHeight * nSrcWidth;
        uint32_t u0 = static_cast<uint32_t>(dpSrc[jUSrc + kUV]);
        
        int jVSrc = tidd * nSrcWidth / 2 + nDstHeight * nSrcWidth * 3 / 2;
        uint32_t v0 = static_cast<uint32_t>(dpSrc[jVSrc + kUV]);
        
        int kDst = tid * 6;
        int jDst = tidd * nDstWidth * 3;

        int r = 1407 * v0 - 720384;
        int g = 716 * v0 + 345 * u0 - 543232;
        int b = 1779 * u0 - 910848;

        uint16_t tt[6];
        tt[0] = (y0 + r) / 1000;
        tt[1] = (y0 - g) / 1000;
        tt[2] = (y0 + b) / 1000;
        tt[3] = (y1 + r) / 1000;
        tt[4] = (y1 - g) / 1000;
        tt[5] = (y1 + b) / 1000;

        tt1[jDst + kDst + 0] = lookupTable[tt[0]];
        tt1[jDst + kDst + 1] = lookupTable[tt[1]];
        tt1[jDst + kDst + 2] = lookupTable[tt[2]];
        tt1[jDst + kDst + 3] = lookupTable[tt[3]];
        tt1[jDst + kDst + 4] = lookupTable[tt[4]];
        tt1[jDst + kDst + 5] = lookupTable[tt[5]];
    }
}