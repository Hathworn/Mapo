#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void resizeBatchKernel(const uint8_t *p_Src, int nSrcPitch, int nSrcHeight, uint8_t *p_dst, int nDstWidth, int nDstHeight) {
    // Compute thread's row and column in output image
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure indices are within bounds
    if (tid < nDstWidth && tidd < nDstHeight) {
        // Calculate scale factors outside of loop
        int yScale = nSrcHeight / nDstHeight;
        int xScale = 3 * (nSrcPitch / nDstWidth);

        // Calculate source and destination indices
        int j = tidd * yScale * nSrcPitch;
        int k = tid * xScale;

        // Read RGB values from source image
        uchar3 rgb;
        rgb.x = p_Src[j + k + 0];
        rgb.y = p_Src[j + k + 1];
        rgb.z = p_Src[j + k + 2];

        // Write RGB values to destination image
        int dstIdx = (tidd * nDstWidth + tid) * 3;
        p_dst[dstIdx + 0] = rgb.x;
        p_dst[dstIdx + 1] = rgb.y;
        p_dst[dstIdx + 2] = rgb.z;
    }
}