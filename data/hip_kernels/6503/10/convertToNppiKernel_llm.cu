#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function for conversion
__global__ static void convertToNppiKernel(uint16_t *dSrc, uint8_t *dDst, int nSrcWidth, int nDstWidth, int nDstHeight, int *lookupTable) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int tidd = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t v0, y0, u0, y2, u1, y1, u2, y3, v1, y5, v2, y4;
    uint16_t tt[6];
    uint4 pF;
    int nDstH = nDstHeight;
    int nDstW = nSrcWidth / 8;

    // Compute only if thread is within valid bounds
    if (tid < nDstW && tidd < nDstH) {
        int j = tidd * nSrcWidth;
        int k = tid * 8;

        // Efficiently load source data
        pF.x = ((uint32_t)dSrc[j + k]) | ((uint32_t)dSrc[j + k + 1] << 16);
        pF.y = ((uint32_t)dSrc[j + k + 2]) | ((uint32_t)dSrc[j + k + 3] << 16);
        pF.z = ((uint32_t)dSrc[j + k + 4]) | ((uint32_t)dSrc[j + k + 5] << 16);
        pF.w = ((uint32_t)dSrc[j + k + 6]) | ((uint32_t)dSrc[j + k + 7] << 16);

        // Efficient unpack for each segment
        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            uint32_t v = (pF.x & 0x3FF00000) >> 20;
            uint32_t y = ((pF.x & 0x000FFC00) >> 10) * 1000;
            uint32_t u = (pF.x & 0x000003FF);

            int r = 1407 * v - 720384;
            int g = 716 * v + 345 * u - 543232;
            int b = 1779 * u - 910848;

            tt[0] = (y + r) / 1000;
            tt[1] = (y - g) / 1000;
            tt[2] = (y + b) / 1000;

            dDst[j * 9 / 4 + (k + i * 6)] = lookupTable[tt[0]];
            dDst[j * 9 / 4 + (k + i * 6 + 1)] = lookupTable[tt[1]];
            dDst[j * 9 / 4 + (k + i * 6 + 2)] = lookupTable[tt[2]];

            tt[3] = ((j + k + i * 2) + r) / 1000;
            tt[4] = ((j + k + i * 2 + 1) - g) / 1000;
            tt[5] = ((j + k + i * 3) + b) / 1000;

            dDst[j * 9 / 4 + (k + i * 6 + 3)] = lookupTable[tt[3]];
            dDst[j * 9 / 4 + (k + i * 6 + 4)] = lookupTable[tt[4]];
            dDst[j * 9 / 4 + (k + i * 6 + 5)] = lookupTable[tt[5]];
        }
    }
}