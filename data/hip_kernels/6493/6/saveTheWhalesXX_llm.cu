#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saveTheWhalesXX (const int d0, const int d1, const int i2, float *xxx, const int d3, const int d4, const float *xx) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    int gridStrideX = blockDim.x * gridDim.x;
    int gridStrideY = blockDim.y * gridDim.y;

    // Use grid-stride loop for better occupancy and scalability
    for (; i < d3; i += gridStrideX) {
        for (; j < d4; j += gridStrideY) {
            xxx[i + j * d0 + i2 * d0 * d1] = xx[i + j * d3];
        }
    }
}