#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void returnQ1(const int dim, const int n, const float *p1, const float *p0, const float *s1, const float *s0, const float *zr, float *q) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        float diff = -0.5f * (s1[i] + p1[i] - s0[i] - p0[i]); // Precompute reused expression
        if (p1[i] == INF || diff < -10.0f) {
            q[i] = 0.0f;
        } else if (diff > 10.0f) {
            q[i] = 1.0e10f; // Use float literal suffix
        } else {
            q[i] = expf(diff) * powf(zr[i], dim - 1); // Use precomputed value
        }
    }
}