#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lhkde(const int n, const float *a, const float *b, float *l, float *h) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        float ai = a[i];      // Load a[i] once
        float bi = 3 * b[i];  // Compute 3 * b[i] once
        l[i] = ai - bi;       // Use stored ai and bi
        h[i] = ai + bi;       // Use stored ai and bi
    }
}