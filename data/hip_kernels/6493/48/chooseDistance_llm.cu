#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void chooseDistance(const int nwl, const int *kex, const float *didi11, float *didi1) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < nwl) {
        // Use register variables for faster access
        int indexWithOffset = i + kex[i] * nwl;
        didi1[i] = __ldg(&didi11[indexWithOffset]); // Use __ldg to utilize read-only cache for didi11
    }
}