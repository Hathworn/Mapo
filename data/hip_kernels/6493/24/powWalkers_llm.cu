#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize by using fast math intrinsic function
__global__ void powWalkers(const int n, const float c, const float *a, float *d) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        d[i] = __powf(a[i], c);  // Use __powf for faster computation
    }
}