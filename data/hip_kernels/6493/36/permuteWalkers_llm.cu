#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void permuteWalkers ( const int dim, const int nwl, const int *kr, const float *xxC, float *xxCP ) {
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure only valid indices are processed
    if ( i < dim && j < nwl ) {
        int t = i + j * dim; // Linear index calculation
        int p = i + kr[j] * dim; // Permuted index calculation
        xxCP[t] = xxC[p]; // Assign permuted value
    }
}