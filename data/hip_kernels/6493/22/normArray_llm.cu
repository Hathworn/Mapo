#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normArray ( const int n, float *a ) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    float c = a[0];

    // Ensure all threads load c after it is set up to provide consistent results
    __shared__ float shared_c;
    if (threadIdx.x == 0) {
        shared_c = c;
    }
    __syncthreads();

    if (i < n) {
        // Use shared memory for repeated global memory access
        a[i] = a[i] / shared_c;
    }
}