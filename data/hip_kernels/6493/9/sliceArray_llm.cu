#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sliceArray(const int n, const int indx, const float *ss, float *zz) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll loop for better performance
    #pragma unroll
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x) {
        zz[idx] = ss[idx + indx];
    }
}