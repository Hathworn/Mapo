#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setDistanceAtLast(const int dim, const int nwl, const float *lst, float *didi) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nwl) {
        // Optimize memory access by reducing calculations inside the loop
        int idx = dim + i * (dim + 4);
        didi[i] = lst[idx];
    }
}