#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sliceIntArray(const int n, const int indx, const int *ss, int *zz) {
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a loop for better load balancing in case n is much larger than blockDim.x
    for (int j = i; j < n; j += blockDim.x * gridDim.x) {
        zz[j] = ss[j + indx];
    }
}