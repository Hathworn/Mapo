#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void returnStatistic ( const int dim, const int nwl, const float *xx, float *s ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Improved bounds checking to avoid redundant threads
    if (i < dim && j < nwl) {
        int t = i + j * dim;
        s[t] = xx[t] * xx[t]; // Eliminating powf for potential performance gain
    }
}