#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shiftWalkers(const int dim, const int nwl, const float *xx, const float *x, float *yy) {
    // Use a single thread index calculation instead of separate i and j calculations
    int index = blockIdx.x * blockDim.x + threadIdx.x + 
                (blockIdx.y * blockDim.y + threadIdx.y) * dim;
    
    // Check both conditions together to reduce branching
    if (threadIdx.x < dim && threadIdx.y < nwl) {
        yy[index] = xx[index] - x[threadIdx.x];
    }
}