#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sortMatrix ( const int nd, const float *a, float *sm ) {
    // Compute row and column indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Compute linear index in the array
    int ij = i + j * nd;

    // Check bounds and perform operation
    if (i < nd && j < nd) {
        sm[ij] = (a[i] > a[j]);
    }
}