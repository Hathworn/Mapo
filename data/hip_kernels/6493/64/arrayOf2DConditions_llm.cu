#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void arrayOf2DConditions(const int dim, const int nwl, const float *bn, const float *xx, float *cc) {
    // Calculate unique thread index
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    // Avoid unnecessary memory access
    if (i < dim && j < nwl) {
        int t = i + j * dim; // Move index computation inside condition
        float x_val = xx[t];
        // Use temporary variables to reduce repetitive computation
        bool cond1 = bn[0 + i * 2] < x_val;
        bool cond2 = x_val < bn[1 + i * 2];
        cc[t] = cond1 * cond2;
    }
}