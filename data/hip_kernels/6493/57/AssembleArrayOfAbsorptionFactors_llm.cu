#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AssembleArrayOfAbsorptionFactors ( const int nmbrOfWlkrs, const int nmbrOfEnrgChnnls, const int nmbrOfElmnts, const float *crssctns, const float *abndncs, const int *atmcNmbrs, const float *wlkrs, float *absrptnFctrs ) {
    int enIndx = threadIdx.x + blockDim.x * blockIdx.x;
    int wlIndx = threadIdx.y + blockDim.y * blockIdx.y;

    if (enIndx < nmbrOfEnrgChnnls && wlIndx < nmbrOfWlkrs) {
        int ttIndx = enIndx + wlIndx * nmbrOfEnrgChnnls;
        float xsctn = 0.0f;

        if (NHINDX == NPRS - 1) {
            float nh = wlkrs[NHINDX + wlIndx * NPRS] * 1.E22;

            // Unroll loop for the first element
            int effElIndx = atmcNmbrs[0] - 1;
            xsctn = abndncs[effElIndx] * crssctns[enIndx * nmbrOfElmnts];

            // Loop through remaining elements
            for (int elIndx = 1; elIndx < nmbrOfElmnts; ++elIndx) {
                effElIndx = atmcNmbrs[elIndx] - 1;
                xsctn += abndncs[effElIndx] * crssctns[elIndx + enIndx * nmbrOfElmnts];
            }
            absrptnFctrs[ttIndx] = expf(-nh * xsctn);
        } else if (NHINDX == NPRS) {
            absrptnFctrs[ttIndx] = 1.0f;
        }
    }
}