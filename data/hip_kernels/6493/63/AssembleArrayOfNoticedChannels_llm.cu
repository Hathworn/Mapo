#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AssembleArrayOfNoticedChannels(const int nmbrOfChnnls, const float lwrNtcdEnrg, const float hghrNtcdEnrg, const float *lwrChnnlBndrs, const float *hghrChnnlBndrs, const float *gdQltChnnls, float *ntcdChnnls) {
    // Use shared memory for faster access and reduction of global memory access
    __shared__ float s_lwrNtcdEnrg;
    __shared__ float s_hghrNtcdEnrg;
    
    if (threadIdx.x == 0) {
        s_lwrNtcdEnrg = lwrNtcdEnrg;
        s_hghrNtcdEnrg = hghrNtcdEnrg;
    }
    __syncthreads();

    // Calculate array index
    int c = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Check bounds only if within range
    if (c < nmbrOfChnnls) {
        float lowerBoundary = lwrChnnlBndrs[c];
        float higherBoundary = hghrChnnlBndrs[c];
        float quality = gdQltChnnls[c];

        // Perform channel notice calculation
        ntcdChnnls[c] = (lowerBoundary > s_lwrNtcdEnrg) * (higherBoundary < s_hghrNtcdEnrg) * (1.0f - quality);
    }
}