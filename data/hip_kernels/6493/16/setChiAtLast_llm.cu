#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setChiAtLast(const int dim, const int nwl, const float *lst, float *stt) {
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Improved access pattern by reducing operations inside loop
    if (i < nwl) {
        int offset = dim + 2 + i * (dim + 4);
        stt[i] = lst[offset];
    }
}