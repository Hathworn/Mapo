#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mapKex(const int nwl, const float *r, int *kex) {
    // Calculate the global thread index
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Optimized boundary check and calculation
    if (i < nwl) {
        // Simplified computation inside kernel
        kex[i] = (int)(r[i] * 2.999999f);  
    }
}