#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void extractParam(const int d, const int n, const int Indx, const float *s, float *a) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride
    while (i < n) { // Use a loop for kernel efficiency
        a[i] = s[Indx + i * d];
        i += stride; // Jump by grid stride
    }
}