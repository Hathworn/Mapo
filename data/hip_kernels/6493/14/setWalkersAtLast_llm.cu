#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setWalkersAtLast (const int dim, const int nwl, const float *lst, float *xx) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Use a constant stride calculation for t
    int stride = dim + 4; // Optimize by pre-computing the stride
    int t = i + j * dim;

    // Use a single if condition to check bounds
    if (i < dim && j < nwl) {
        xx[t] = lst[i + j * stride];
    }
}