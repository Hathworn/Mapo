#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void arrayOfPriors1 (const int dim, const int nwl, const float *cn, const float *nhMd, const float *nhSg, const float *xx, float *pr) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < nwl) {
        // Removing unnecessary computations and simplifying logic
        float sum = 0.0f; 
        bool condition = (cn[i] == dim);
        pr[i] = condition * sum + (!condition) * INF;
    }
}