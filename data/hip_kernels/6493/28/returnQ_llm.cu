#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void returnQ (const int dim, const int n, const float *s1, const float *s0, const float *zr, float *q) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for loop unrolling
    for (; i < n; i += stride) { // Use loop to handle more data per thread
        float diff = s1[i] - s0[i]; // Precompute s1[i] - s0[i]
        q[i] = expf(-0.5f * diff) * powf(zr[i], dim - 1); // Compute once using precomputed value
    }
}