#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sortIndexKde(const int d, const int n, const float *a, const float *b, float *sa, float *sb) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    // Early exit if index out of bounds
    if (i >= d || j >= n) return;

    int ij = i + j * d;
    float mewa = a[ij];
    float mewb = b[ij];
    int mewj = j;
    
    for (int l = 0; l < n; l++) {
        int il = i + l * d;
        float ail = a[il];
        float bil = b[il];

        // Use ternary operators for cleaner conditional updates
        bool greater = (ail > mewa);
        bool lesser = (ail < mewa);
        
        mewj += greater * (l - mewj) - lesser * (mewj - l);
        mewa += greater * (ail - mewa) + lesser * (ail - mewa);
        mewb += greater * (bil - mewb) + lesser * (bil - mewb);
    }
    
    sa[ij] = mewa;
    sb[ij] = mewb;
}