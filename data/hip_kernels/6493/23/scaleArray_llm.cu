#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleArray(const int n, const float c, float* __restrict__ a) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        // Using __restrict__ to potentially increase memory access efficiency
        a[i] *= c; // Simplified the multiplication operation
    }
}