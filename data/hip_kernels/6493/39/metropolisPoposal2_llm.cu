#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void metropolisPoposal2 (const int dim, const int nwl, const int isb, const float *xx, const float *rr, float *xx1) {
    // Use a 1D grid and calculate a 2D index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate flat index
    if (i < dim && j < nwl) {
        int t = i + j * dim;
        // Simplify conditional addition
        xx1[t] = xx[t] + (i == isb ? rr[j] : 0.0f);
    }
}