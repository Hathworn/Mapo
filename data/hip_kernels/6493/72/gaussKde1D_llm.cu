#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gaussKde1D ( const int dim, const int nd, const int nb, const int Indx, const float *hh, const float *a, const float *b, float *pdf ) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;
    if ( i < nb && j < nd ) {  // Ensure index bounds
        int ij = i + j * nb;
        float h = hh[Indx];
        float diff = a[Indx + j * dim] - b[Indx + i * dim];  // Precompute difference
        float expTerm = expf(-0.5 * (diff * diff) / (h * h));  // Simplify calculation
        pdf[ij] = expTerm / (h * sqrtf(2 * PI));  // Simplify denominator
    }
}