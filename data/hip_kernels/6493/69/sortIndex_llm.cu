#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sortIndex(const int d, const int n, const float *a, int *si, float *sa) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    if (i < d && j < n) { // Check thread is in range
        int ij = i + j * d;
        int mewj = j;
        float mewa = a[ij];

        // Optimize loop by unrolling
        #pragma unroll
        for (int l = 0; l < n; l++) {
            int il = i + l * d;
            float value = a[il];

            // Use ternary operator for compactness
            if (l != j) {
                bool condition = (l > j) ? (value < mewa) : (value > mewa);
                mewj += condition * (l - mewj);
                mewa += condition * (value - mewa);
            }
        }

        si[ij] = mewj; // Assign sorted index
        sa[ij] = mewa; // Assign sorted value
    }
}