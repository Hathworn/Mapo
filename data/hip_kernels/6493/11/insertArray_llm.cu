#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void insertArray(const int n, const int indx, const float *ss, float *zz) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // Unroll loop to optimize memory accesses
    #pragma unroll
    for (; i < n; i += blockDim.x * gridDim.x) {
        zz[indx + i] = ss[i];
    }
}