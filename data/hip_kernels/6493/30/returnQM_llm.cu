#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void returnQM ( const int dim, const int n, const float *s1, const float *s0, float *q ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop unrolling for improved performance
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x) {
        float diff = s1[idx] - s0[idx]; // Store repeated subexpression
        q[idx] = expf(-0.5f * diff); // Compute once per iteration
    }
}