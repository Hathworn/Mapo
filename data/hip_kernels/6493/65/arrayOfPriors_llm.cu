#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void arrayOfPriors ( const int dim, const int nwl, const float *cn, const float *xx, float *pr ) {
    // Calculate the global index once to avoid redundant computations
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nwl) {
        // Directly assign the result instead of using multiplication and addition
        pr[i] = (cn[i] == dim) ? 0.0f : INF;
    }
}