#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void chooseLaw ( const int nwl, const int *kex, const float *didi11, const float *didi12, const float *didi13, float *didi1 ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if ( i < nwl ) {
        int choice = kex[i]; // Cache kex value
        if(choice == 0) {
            didi1[i] = didi11[i]; // Select didi1 based on the kex value
        } else if(choice == 1) {
            didi1[i] = didi12[i];
        } else if(choice == 2) {
            didi1[i] = didi13[i];
        }
    } 
}