#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void chainFunction ( const int dim, const int nwl, const int nst, const int ipr, const float *smpls, float *chnFnctn ) {
    // Calculate the linear index for the current thread
    int t = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * nwl;
    
    // Ensure within bounds before accessing memory
    if (threadIdx.x + blockIdx.x * blockDim.x < nwl && threadIdx.y + blockIdx.y * blockDim.y < nst) {
        chnFnctn[t] = smpls[ipr + t * dim];
    }
}