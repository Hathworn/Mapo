#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setPriorAtLast(const int dim, const int nwl, const float *lst, float *prr) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate grid stride.
    for (; i < nwl; i += stride) {       // Use grid stride loop to handle more elements.
        prr[i] = lst[dim + 3 + i * (dim + 4)]; // Compute position in `lst` array.
    }
}