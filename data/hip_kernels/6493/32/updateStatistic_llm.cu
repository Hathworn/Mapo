#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateStatistic ( const int nwl, const float *stt1, const float *q, const float *r, float *stt0 ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if ( i < nwl ) {
        // Use ternary operator for concise conditional assignment
        stt0[i] = (q[i] > r[i]) ? stt1[i] : stt0[i];
    }
}