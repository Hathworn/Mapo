#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scaleWalkers ( const int n, const float c, const float *a, float *d ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for optimization
    for (; i < n; i += stride) { // Use strided loop for better memory access
        d[i] = c * a[i]; // Perform scaling
    }
}