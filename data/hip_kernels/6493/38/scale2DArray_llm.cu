#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale2DArray ( const int dim, const int nwl, const float *zr, const float *xx, float *xx1 ) {
    // Calculate flat 1D global index for the 2D grid
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    // Ensure index is within bounds before accessing arrays
    if (i < dim && j < nwl) {
        int t = i + j * dim; // Compute index only if within bounds
        xx1[t] = zr[j] * xx[t]; // Perform scaling
    }
}