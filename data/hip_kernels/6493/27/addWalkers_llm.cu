#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addWalkers ( const int dim, const int nwl, const float *xx0, const float *xxW, float *xx1 ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Check boundaries before computation to avoid unnecessary work
    if (i < dim && j < nwl) {
        int t = i + j * dim; // Calculate once, use multiple times
        xx1[t] = __fmaf_rn(xx0[t], 1.0f, xxW[t]);  // Use fused multiply-add for better performance
    }
}