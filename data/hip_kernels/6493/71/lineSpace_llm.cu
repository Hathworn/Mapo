#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lineSpace (const int d, const int n, const float *l, const float *h, float *b) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate delta only once per thread block
    __shared__ float sharedDelta[256]; // Assuming max 256 threads per block for simplicity
    if (j == 0 && i < d) {
        sharedDelta[threadIdx.x] = (h[i] - l[i]) / (n - 1);
    }
    __syncthreads(); // Wait for all threads to load delta

    if (i < d && j < n) {
        b[i + j * d] = l[i] + j * sharedDelta[threadIdx.x];
    }
}