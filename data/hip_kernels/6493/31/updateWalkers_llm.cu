#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateWalkers (const int dim, const int nwl, const float *xx1, const float *q, const float *r, float *xx0) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Optimize boundary check and ensure valid memory access
    if (i < dim && j < nwl) {
        int t = i + j * dim; 
        // Use conditional operator to avoid redundant branches
        xx0[t] = (q[j] > r[j]) * xx1[t] + (q[j] <= r[j]) * xx0[t];
    }
}