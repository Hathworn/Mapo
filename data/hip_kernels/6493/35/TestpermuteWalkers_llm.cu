#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TestpermuteWalkers ( const int dim, const int nwl, const int *kr, const float *xxC, float *xxCP ) {
    // Calculate global thread index in one dimension
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;  // Calculate stride for loop

    // Use a loop to iterate over elements, reducing grid size needed
    for (int t = idx; t < dim * nwl; t += stride) {
        int i = t % dim;  // Calculate 2D indices from 1D index
        int j = t / dim;
        if (i < dim && j < nwl) {
            xxCP[t] = xxC[t];  // Directly map input to output
        }
    }
}