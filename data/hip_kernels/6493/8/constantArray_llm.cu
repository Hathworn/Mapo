#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constantArray(const int n, const float c, float *a) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    // Use loop unrolling to optimize memory operations
    int stride = blockDim.x * gridDim.x;
    for (; i < n; i += stride) {
        a[i] = c;
    }
}