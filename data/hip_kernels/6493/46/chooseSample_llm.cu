#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void chooseSample (const int nDB, const int si, const float *EE, float *EBV) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Efficient memory access with appropriate bounds checking
    if (i < nDB) {
        EBV[i] = EE[i + si * nDB];
    }
}