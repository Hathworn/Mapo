#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initializeAtRandom(const int dim, const int nwl, const float dlt, const float *x0, const float *stn, float *xx) {
    // Calculate 1D global thread index
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (t < dim * nwl) {
        // Calculate linear indices for x0 and stn
        int i = t % dim;
        xx[t] = x0[i] + dlt * stn[t];
    }
}