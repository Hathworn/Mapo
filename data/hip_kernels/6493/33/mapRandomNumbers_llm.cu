#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mapRandomNumbers(const int nwl, const int ist, const int isb, const float *r, float *zr, int *kr, float *ru, int *kex) {
    // Calculate the global thread index
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int rr_base = isb * 4 * nwl + ist * 4 * 2 * nwl; // Precompute base offset for rr
    
    if (i < nwl) {
        int rr = i + rr_base;
        // Unroll loop iterations for better performance
        zr[i] = 1.0f / ACONST * powf(r[rr] * (ACONST - 1.0f) + 1.0f, 2.0f);
        rr += nwl;
        kr[i] = (int)truncf(r[rr] * (nwl - 0.000001f));
        rr += nwl;
        ru[i] = r[rr];
        rr += nwl;
        kex[i] = (int)truncf(r[rr] * 4.999999f); // Simplified constant
    }
}