#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void substractWalkers(const int dim, const int nwl, const float *xx0, const float *xxCP, float *xx1) {
    // Calculate 1D index for the grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (idx < dim * nwl) {
        xx1[idx] = xx0[idx] - xxCP[idx];
    }
}