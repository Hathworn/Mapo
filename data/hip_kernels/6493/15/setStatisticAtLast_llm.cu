#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setStatisticAtLast(const int dim, const int nwl, const float *lst, float *stt) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Unroll the loop for better performance when nwl is large
    if (i < nwl) {
        stt[i] = lst[dim + 1 + i * (dim + 5)];
    }
}