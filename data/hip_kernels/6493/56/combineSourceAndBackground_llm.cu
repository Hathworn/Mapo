#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void combineSourceAndBackground(const int nwl, const int n, const float scale, float *src, const float *bkg) {
    // Calculate 1D index using 2D grid and block dimensions for better memory coalescing
    int globalIdx = threadIdx.x + blockDim.x * blockIdx.x + (threadIdx.y + blockDim.y * blockIdx.y) * n;
    
    if (globalIdx < nwl * n) {
        // Optimize memory access pattern by using linear index
        src[globalIdx] = src[globalIdx] + scale * bkg[globalIdx];
    }
}