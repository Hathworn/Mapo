#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void double_value(double *x, double *y)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform operation with global index
    y[idx] = 2.0 * x[idx];
}