#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Return early if out of bounds
    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Compute target pixel coordinates
    float cx = __fmaf_rn(u_, time_scale, (float)j + 1.0f);
    float cy = __fmaf_rn(v_, time_scale, (float)i + 1.0f);
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    float value = src[image_row_offset + j];

    // Check bounds and update pixel with atomic add
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}