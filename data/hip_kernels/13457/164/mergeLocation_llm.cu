#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* __restrict__ loc_, float* __restrict__ x, float* __restrict__ y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure that we do not access out-of-bounds memory
    if (ptidx >= npoints) return;

    short2 loc = loc_[ptidx];

    // Minimize memory latency by using temporary variables
    float loc_x = loc.x * scale;
    float loc_y = loc.y * scale;

    x[ptidx] = loc_x;
    y[ptidx] = loc_y;
}