#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x; // Compute global thread index
    hiprand_init(seed, tidx, 0, &state[tidx]); // Initialize each thread's hiprand state
}