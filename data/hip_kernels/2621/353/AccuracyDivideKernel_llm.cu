#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Optimize by using shared memory to reduce global memory access
    extern __shared__ float shared_accuracy[];
    if (threadIdx.x == 0) {
        shared_accuracy[0] = *accuracy;
    }
    __syncthreads();
    
    // Single thread update to avoid race condition
    if (threadIdx.x == 0) {
        shared_accuracy[0] /= N;
        *accuracy = shared_accuracy[0];
    }
}