#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        sum += a[eidx] * b[eidx]; // Accumulate dot product in register
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Use loop unrolling for reduction within warp
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (threadIdx.x < s) {
            shmem[threadIdx.x] += shmem[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}