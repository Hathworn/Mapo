#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];
        
        // Compute log probability
        labelLogProbs[tx] = __logf(labelp);

        // Check if prediction matches max probability
        if (labelp != maxp) {
            // Assign zero probability if no match
            correctProbs[tx] = 0.0f;
        } else {
            // Count labels with max probability
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp);
            }
            // Assign correctness probability
            correctProbs[tx] = 1.0f / static_cast<float>(numMax);
        }
    }
}