#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Compute a unique global index for each thread
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    // Loop through elements more efficiently using thread stride
    for (uint i = idx; i < tgtWidth * tgtHeight; i += stride) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        
        // Calculate source positions using modulo to correctly tile
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Assign tiled values to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}