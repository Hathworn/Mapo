#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void refreshClusters(dim3 *sum, dim3 *cluster, int *counter)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure only valid thread indices access memory
    if(i < gridDim.x * blockDim.x) {
        int count = counter[i];

        // Use a single branch for better warp execution
        if (count != 0) {
            cluster[i].x = sum[i].x / count;
            cluster[i].y = sum[i].y / count;
            cluster[i].z = sum[i].z / count;
        } else {
            cluster[i] = dim3(0, 0, 0);
        }

        // Reset sum and counter values
        sum[i] = dim3(0, 0, 0);
        counter[i] = 0;
    }
}