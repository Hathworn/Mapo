#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;

    extern __shared__ float shared_buffer[]; // Use dynamically allocated shared memory
    float local_sum = 0.0f;

    // Compute partial sum for dot product using warp-level reduction
    for (int i = idx; i < dim; i += blockDim.x) {
        local_sum += dY[i] * Y[i];
    }
    shared_buffer[idx] = local_sum;
    __syncthreads();

    // Use parallel reduction for higher efficiency
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (idx < stride) {
            shared_buffer[idx] += shared_buffer[idx + stride];
        }
        __syncthreads();
    }

    // Compute gradient
    const float dot_product = shared_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - dot_product); // Efficient direct calculation
    }
}