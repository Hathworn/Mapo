#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Ensure only one thread performs the division for global memory write
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}