#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CudaPermutePVToCudnn( float * __restrict__ dest, float * __restrict__ src, int outFeatures, int ny, int nx, int inFeatures, int manyScaleX, int manyScaleY, int cropX, int cropY) {
    // compute destination parameters using constants to avoid repetitive operations
    const int destNx = (nx - 2 * cropX) / manyScaleX;
    const int destNy = (ny - 2 * cropY) / manyScaleY;
    const int destInFeatures = inFeatures * manyScaleX * manyScaleY;
    
    // calculate source index
    int kSrc = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (kSrc < outFeatures * ny * nx * inFeatures) {
        int kOF = kSrc / (ny * nx * inFeatures);
        int kY  = (kSrc % (ny * nx * inFeatures)) / (nx * inFeatures);
        int kX  = (kSrc % (nx * inFeatures)) / inFeatures;
        int kIF = kSrc % inFeatures;
        
        // check if indices are within bounds and adjust
        if (kX >= cropX && kX < nx - cropX && kY >= cropY && kY < ny - cropY) {
            kX -= cropX;
            kY -= cropY;
            
            // compute new feature index and scaled positions
            kIF += inFeatures * (kX % manyScaleX + (kY % manyScaleY) * manyScaleX);
            kX /= manyScaleX;
            kY /= manyScaleY;
            
            // calculate flattened destination index
            const int sOF = destInFeatures * destNy * destNx;
            const int sIF = destNy * destNx;
            const int sY  = destNx;
            int kDest = kOF * sOF + kIF * sIF + kY * sY + kX;
            
            // populate destination array
            dest[kDest] = src[kSrc];
        }
    }
}