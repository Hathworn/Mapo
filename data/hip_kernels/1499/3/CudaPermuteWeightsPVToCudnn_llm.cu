#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CudaPermuteWeightsPVToCudnn(float *dest, float *src, int outFeatures, int ny, int nx, int inFeatures, int manyScaleX, int manyScaleY) {
    // Calculate global thread ID for flattened 3D grid
    int kSrc = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    // Stride for parallel computation
    int totalElements = outFeatures * manyScaleX * manyScaleY * ny * nx * inFeatures;
    
    // Ensure that the thread is within bounds
    for (int i = kSrc; i < totalElements; i += blockDim.x * gridDim.x) {
        int kOF = i / (ny * nx * inFeatures);
        int kY  = (i % (ny * nx * inFeatures)) / (nx * inFeatures);
        int kX  = (i % (nx * inFeatures)) / inFeatures;
        int kIF = (i % inFeatures);
        
        int sOF = inFeatures * ny * nx;
        int sIF = ny * nx;
        int sY  = nx;

        // Calculate destination index with transformed order
        int kDest = kOF * sOF + kIF * sIF + (ny - kY - 1) * sY + (nx - kX - 1);

        dest[kDest] = src[i];
    }
}