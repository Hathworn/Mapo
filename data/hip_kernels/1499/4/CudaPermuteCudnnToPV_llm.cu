#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CudaPermuteCudnnToPV(float *dest, float *src, int outFeatures, int ny, int nx, int inFeatures, int manyScaleX, int manyScaleY) {
    int srcNx = nx / manyScaleX;
    int srcNy = ny / manyScaleY;
    int srcInFeatures = inFeatures * manyScaleX * manyScaleY;

    int kDest = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a single fetch if the index is within bounds
    if (kDest >= outFeatures * ny * nx * inFeatures) return;

    int kOF = kDest / (ny * nx * inFeatures);
    int remaining = kDest % (ny * nx * inFeatures);
    int kY = remaining / (nx * inFeatures);
    remaining = remaining % (nx * inFeatures);
    int kX = remaining / inFeatures;
    int kIF = remaining % inFeatures;

    // Update indices based on manyScale values
    kIF += inFeatures * (kX % manyScaleX + (kY % manyScaleY) * manyScaleX);
    kX /= manyScaleX;
    kY /= manyScaleY;

    int sOF = srcInFeatures * srcNy * srcNx;
    int sIF = srcNy * srcNx;
    int sY = srcNx;

    int kSrc = kOF * sOF + kIF * sIF + kY * sY + kX;

    dest[kDest] = src[kSrc];
}