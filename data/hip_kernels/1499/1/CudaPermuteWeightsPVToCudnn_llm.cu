#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void CudaPermuteWeightsPVToCudnn(float *dest, float *src, int numArbors, int outFeatures, int ny, int nx, int inFeatures) {
    // Calculate the global thread index
    int kSrc = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if the thread is within bounds
    if (kSrc < outFeatures * ny * nx * inFeatures) {
        int kA = kSrc / (outFeatures * ny * nx * inFeatures);
        int remaining = kSrc % (outFeatures * ny * nx * inFeatures);

        int kOF = remaining / (ny * nx * inFeatures);
        remaining = remaining % (ny * nx * inFeatures);

        int kY = remaining / (nx * inFeatures);
        int kX = (remaining % (nx * inFeatures)) / inFeatures;
        int kIF = remaining % inFeatures;

        // Calculate destination index
        int kDest = kA * outFeatures * inFeatures * ny * nx
            + kOF * inFeatures * ny * nx
            + kIF * ny * nx
            + (ny - kY - 1) * nx
            + (nx - kX - 1);
        
        dest[kDest] = src[kSrc]; // Assign the permuted value to destination
    }
}