#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef is_CUDA
#endif

__global__ void testkernel()
{
    double p = threadIdx.x + 66;
    // Optimization: Use shared memory to cache sqrt values to reduce redundant calculation
    __shared__ double sqrt_cache[256]; // Assume max 256 threads per block
    sqrt_cache[threadIdx.x] = std::sqrt(p);

    // Unroll loop to reduce overhead
    for (int i = 0; i < 30000000; i += 5)
    {
        p += i / p - sqrt_cache[threadIdx.x];
        p += (i + 1) / p - sqrt_cache[threadIdx.x];
        p += (i + 2) / p - sqrt_cache[threadIdx.x];
        p += (i + 3) / p - sqrt_cache[threadIdx.x];
        p += (i + 4) / p - sqrt_cache[threadIdx.x];
    }

    // Reduced printf frequency to minimize performance impact
    if(threadIdx.x == 0 && blockIdx.x == 0)
    {
        printf("thread %d; block %d\n", threadIdx.x, blockIdx.x);
    }
}