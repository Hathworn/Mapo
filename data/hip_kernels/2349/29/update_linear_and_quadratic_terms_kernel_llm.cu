#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    // Using local variables to avoid repeated global access
    float cur_weight = *cur_tot_weight;
    float new_num_frames = old_num_frames + cur_weight;
    float prior_scale_change = 1.0f;

    // Avoid checking non-zero with floating point
    if(max_count > 0) {
        float old_prior_scale = max(old_num_frames, (float)max_count) / max_count;
        float new_prior_scale = max(new_num_frames, (float)max_count) / max_count;
        prior_scale_change += new_prior_scale - old_prior_scale;
    }

    // Optimize loop using built-in device computation
    for (int32_t idx = blockIdx.x * blockDim.x + threadIdx.x; idx < n; idx += blockDim.x * gridDim.x) {
        int32_t diag_idx = ((idx + 1) * (idx + 2) / 2) - 1;
        atomicAdd(&quadratic[diag_idx], prior_scale_change); // Ensure atomic operation for data integrity
    }

    // Reduce branching, use atomic operation for linear update
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        atomicAdd(&linear[0], prior_offset * prior_scale_change); // Ensure atomic operation for data integrity
    }
}