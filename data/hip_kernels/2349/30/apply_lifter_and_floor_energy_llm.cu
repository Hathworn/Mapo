#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    float *feats = features + frame * ldf;

    // Apply lifter coefficients using shared memory for coefficients
    __shared__ float shared_lifter_coeffs[1024];  // Assuming max num_cols <= 1024
    if (thread_id < num_cols) {
        shared_lifter_coeffs[thread_id] = lifter_coeffs[thread_id];
    }
    __syncthreads();

    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += blockDim.x) {
            float lift = shared_lifter_coeffs[c];
            float f = feats[c];
            feats[c] = f * lift;
        }
    }

    // Only use the first thread in a block to handle energy flooring
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = log(energy_floor);

        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        feats[0] = energy;
    }
}