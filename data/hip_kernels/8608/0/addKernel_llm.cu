#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize by checking bounds
    if (i < gridDim.x * blockDim.x) {
        c[i] = a[i] + b[i];
    }
}