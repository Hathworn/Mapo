#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void checkIndex(void) {
    // Using a single call to retrieve block/thread and grid dimensions
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    printf("Thread ID: %d -- threadIdx:(%d, %d, %d) blockIdx:(%d, %d, %d) blockDim:(%d, %d, %d) gridDim:(%d, %d, %d)\n", 
           tid,
           threadIdx.x, threadIdx.y, threadIdx.z,
           blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z,
           gridDim.x, gridDim.y, gridDim.z);
}