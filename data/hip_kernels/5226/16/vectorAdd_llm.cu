#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = hipBlockDim_x * hipBlockIdx_x + hipThreadIdx_x;

    // Use built-in variable for efficiency, handling boundary check.
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
    // Removed unnecessary printf for cleaner execution.
}