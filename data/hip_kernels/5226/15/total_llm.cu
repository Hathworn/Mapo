#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void total(float *input, float *output, int len) {
    __shared__ float partialSum[2 * BLOCK_SIZE];
    unsigned int t = threadIdx.x, start = 2 * blockIdx.x * BLOCK_SIZE;

    // Load elements into shared memory, using a single conditional
    partialSum[t] = (start + t < len) ? input[start + t] : 0;
    partialSum[BLOCK_SIZE + t] = (start + BLOCK_SIZE + t < len) ? input[start + BLOCK_SIZE + t] : 0;
    __syncthreads();

    // Reduce the shared memory array
    for (unsigned int stride = BLOCK_SIZE; stride >= 1; stride >>= 1) {
        if (t < stride) {
            partialSum[t] += partialSum[t + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (t == 0) {
        output[blockIdx.x] = partialSum[0];
    }
}