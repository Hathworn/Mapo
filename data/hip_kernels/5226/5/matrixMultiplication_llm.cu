#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define tileSize 32

//function for data initialization
void initialization( double *M,  double *N, int arow, int acol, int brow, int bcol);
//(for Debugging) prints out the input data
void printInput( double *M,  double *N, int arow, int acol, int brow,  int bcol);
//(for Debugging) prints out the output data
void printOutput( double *P_C,  double *P_G, int arow, int bcol);

//GPU kernel
__global__ void matrixMultiplication(double* M, double* N, double* P, int widthAHeightB, int heightA, int widthB) {
    // Calculate block-specific matrix starting points
    int Mstart = widthAHeightB * tileSize * blockIdx.y;
    int Nstart = tileSize * blockIdx.x;
    double temp = 0;

    // Allocate shared memory for tiles
    __shared__ double Ms[tileSize][tileSize];
    __shared__ double Ns[tileSize][tileSize];

    for (int m = Mstart, n = Nstart; m < Mstart + widthAHeightB; m += tileSize, n += tileSize * widthB) {
        // Load tiles into shared memory with bounds checking
        if (m + widthAHeightB * threadIdx.y + threadIdx.x < widthAHeightB * heightA) {
            Ms[threadIdx.y][threadIdx.x] = M[m + widthAHeightB * threadIdx.y + threadIdx.x];
        } else {
            Ms[threadIdx.y][threadIdx.x] = 0.0;
        }

        if (n + widthB * threadIdx.y + threadIdx.x < widthAHeightB * widthB) {
            Ns[threadIdx.y][threadIdx.x] = N[n + widthB * threadIdx.y + threadIdx.x];
        } else {
            Ns[threadIdx.y][threadIdx.x] = 0.0;
        }
        __syncthreads();

        // Perform computation on the tiles
        for (int i = 0; i < tileSize; ++i) {
            temp += Ms[threadIdx.y][i] * Ns[i][threadIdx.x];
        }
        __syncthreads();
    }

    // Write the computed value to the output matrix
    int row = blockIdx.y * tileSize + threadIdx.y;
    int col = blockIdx.x * tileSize + threadIdx.x;
    if (row < heightA && col < widthB) {
        P[widthB * row + col] = temp;
    }
}