#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulVarSharedMemoryKernel(float* M, float* N, float* P, int widthAHeightB, int heightA, int widthB) {

    int Mstart=widthAHeightB*tileSize*blockIdx.y;
    int Mend=Mstart+ widthAHeightB - 1;
    int mstep=tileSize;
    int Nstart=tileSize*blockIdx.x;
    int nstep=tileSize*widthB;
    float temp=0;

    __shared__ float Ms[tileSize][tileSize];
    __shared__ float Ns[tileSize][tileSize];
    
    // Simplified area where the tiles fit without boundary issues
    for(int m = Mstart, n = Nstart; m <= Mend; m += mstep, n += nstep) {
        
        // Load shared memory with conditions to prevent out-of-bounds
        Ms[threadIdx.y][threadIdx.x] = (m % widthAHeightB + threadIdx.x < widthAHeightB && blockIdx.y * tileSize + threadIdx.y < heightA) ? 
                                       M[m + widthAHeightB * threadIdx.y + threadIdx.x] : 0.0f;
        
        Ns[threadIdx.y][threadIdx.x] = ((n / widthB) + threadIdx.y < widthAHeightB && blockIdx.x * tileSize + threadIdx.x < widthB) ? 
                                       N[n + widthB * threadIdx.y + threadIdx.x] : 0.0f;

        __syncthreads();
        
        // Compute partial results
        for (int i = 0; i < tileSize; ++i) {
            temp += Ms[threadIdx.y][i] * Ns[i][threadIdx.x];
        }
        __syncthreads();
    }
    
    // Write the result to global memory with bounds checking
    if(blockIdx.y * tileSize + threadIdx.y < heightA && blockIdx.x * tileSize + threadIdx.x < widthB) {
        P[widthB * tileSize * blockIdx.y + tileSize * blockIdx.x + widthB * threadIdx.y + threadIdx.x] = temp;
    }
}