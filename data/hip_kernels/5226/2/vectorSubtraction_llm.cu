#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define tileSize 32

//function for data initialization
void initialization( double *M,  double *N, int arow, int acol, int brow, int bcol);
//(for Debugging) prints out the input data
void printInput( double *M,  double *N, int arow, int acol, int brow,  int bcol);
//(for Debugging) prints out the output data
void printOutput( double *P_C,  double *P_G, int arow, int bcol);

//GPU kernels

__global__ void vectorSubtraction(const double *A, const double *B, double *C, int numElements)
{
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (index < numElements) { // Ensure index is within bounds
        C[index] = A[index] - B[index];
    }
}