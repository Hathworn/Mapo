#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define tileSize 32

//function for data initialization
void initialization( double *M,  double *N, int arow, int acol, int brow, int bcol);
//(for Debugging) prints out the input data
void printInput( double *M,  double *N, int arow, int acol, int brow,  int bcol);
//(for Debugging) prints out the output data
void printOutput( double *P_C,  double *P_G, int arow, int bcol);

//GPU kernels

__global__ void vectorScaling(const double *A, double s, double *C, int numElements)
{
    // Calculate the global index for the current thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we do not exceed bounds of the array
    if (index < numElements)
    {
        C[index] = A[index] * s;  // Perform the scaling operation
    }
}