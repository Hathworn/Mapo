#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width) {
    // Cache the row and column into registers to reduce memory access
    int Row = blockIdx.y * blockDim.y + threadIdx.y; // Calculate the row index
    int Col = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the column index

    // Ensure row and column are within matrix bounds
    if (Row < Width && Col < Width) {
        float Pvalue = 0;

        // Use shared memory for input matrices to optimize memory access
        __shared__ float sharedM[TILE_WIDTH][TILE_WIDTH];
        __shared__ float sharedN[TILE_WIDTH][TILE_WIDTH];

        for (int k = 0; k < (Width + TILE_WIDTH - 1) / TILE_WIDTH; ++k) {
            if (Row < Width && k * TILE_WIDTH + threadIdx.x < Width)
                sharedM[threadIdx.y][threadIdx.x] = M[Row * Width + k * TILE_WIDTH + threadIdx.x];
            else
                sharedM[threadIdx.y][threadIdx.x] = 0.0;

            if (Col < Width && k * TILE_WIDTH + threadIdx.y < Width)
                sharedN[threadIdx.y][threadIdx.x] = N[(k * TILE_WIDTH + threadIdx.y) * Width + Col];
            else
                sharedN[threadIdx.y][threadIdx.x] = 0.0;

            __syncthreads();

            for (int j = 0; j < TILE_WIDTH; ++j)
                Pvalue += sharedM[threadIdx.y][j] * sharedN[j][threadIdx.x];

            __syncthreads();
        }

        // Write the result to the output matrix P
        P[Row * Width + Col] = Pvalue;
    }
}
```
