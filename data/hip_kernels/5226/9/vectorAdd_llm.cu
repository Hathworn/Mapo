#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access by using shared memory
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory for loading data
    __shared__ float sharedA[256];
    __shared__ float sharedB[256];

    if (i < numElements)
    {
        sharedA[threadIdx.x] = A[i];
        sharedB[threadIdx.x] = B[i];
        __syncthreads();  // Ensure all data is loaded before proceeding

        C[i] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
    }
}