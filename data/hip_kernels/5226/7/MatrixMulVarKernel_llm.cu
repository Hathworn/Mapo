#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulVarKernel(float* M, float* N, float* P, int widthAHeightB, int heightA, int widthB) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;  // Calculate the row index of the P element and M
    int Col = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate the column index of P and N
    if (Row < heightA && Col < widthB) {
        float Pvalue = 0;
        // Loop unrolling for better performance by processing two elements per iteration
        for (int k = 0; k < widthAHeightB - 1; k += 2) {
            Pvalue += M[Row * widthAHeightB + k] * N[k * widthB + Col];
            Pvalue += M[Row * widthAHeightB + k + 1] * N[(k + 1) * widthB + Col];
        }
        // Process any remaining element outside the loop if widthAHeightB is odd
        if (widthAHeightB % 2 != 0) {
            Pvalue += M[Row * widthAHeightB + (widthAHeightB - 1)] * N[(widthAHeightB - 1) * widthB + Col];
        }
        P[Row * widthB + Col] = Pvalue;
    }
}