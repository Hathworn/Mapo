#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    // Use shared memory for improved performance
    extern __shared__ float sharedData[];

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int t = threadIdx.x;

    // Load data into shared memory
    if (i < numElements) {
        sharedData[t * 2] = A[i];
        sharedData[t * 2 + 1] = B[i];
    }
    __syncthreads();

    // Perform vector addition using shared memory
    if (i < numElements) {
        C[i] = sharedData[t * 2] + sharedData[t * 2 + 1];
    }
}