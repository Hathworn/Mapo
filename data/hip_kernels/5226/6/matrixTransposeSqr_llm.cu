#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define tileSize 32

//function for data initialization
void initialization(double *M, double *N, int arow, int acol, int brow, int bcol);
//(for Debugging) prints out the input data
void printInput(double *M, double *N, int arow, int acol, int brow, int bcol);
//(for Debugging) prints out the output data
void printOutput(double *P_C, double *P_G, int arow, int bcol);

// Optimized GPU kernel
__global__ void matrixTransposeSqr(double *P, double *M, int width, int height)
{
    // Use shared memory to optimize memory access
    __shared__ double tile[tileSize][tileSize + 1]; // Avoid bank conflicts

    // Calculate indices
    unsigned int xIdx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int yIdx = blockDim.y * blockIdx.y + threadIdx.y;

    // Load data into shared memory
    if (xIdx < width && yIdx < height)
    {
        unsigned int inIdx = xIdx + width * yIdx;
        tile[threadIdx.y][threadIdx.x] = M[inIdx];
    }

    __syncthreads();

    // Transpose and store data back to global memory
    xIdx = blockDim.y * blockIdx.y + threadIdx.x;
    yIdx = blockDim.x * blockIdx.x + threadIdx.y;

    if (xIdx < height && yIdx < width)
    {
        unsigned int outIdx = xIdx + height * yIdx;
        P[outIdx] = tile[threadIdx.x][threadIdx.y];
    }
}