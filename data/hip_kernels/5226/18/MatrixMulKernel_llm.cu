#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK_SIZE 16

__global__ void MatrixMulKernel(float* M, float* N, float* P, int Width) {
    // Calculate tile starting points and steps
    int Mstart = Width * BLOCK_SIZE * blockIdx.y;
    int Nstart = BLOCK_SIZE * blockIdx.x;
    int mstep = BLOCK_SIZE * Width;
    int nstep = BLOCK_SIZE;
    float temp = 0;

    for (int m = Mstart, n = Nstart; m < Mstart + Width; m += mstep, n += nstep) {
        __shared__ float Ms[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Ns[BLOCK_SIZE][BLOCK_SIZE];

        // Load data into shared memory avoiding bank conflicts
        int row = threadIdx.y;
        int col = threadIdx.x;
        Ms[row][col] = M[m + Width * row + col];
        Ns[row][col] = N[n + Width * row + col];
        __syncthreads();

        // Perform multiplication using shared memory
        for (int i = 0; i < BLOCK_SIZE; ++i) {
            temp += Ms[row][i] * Ns[i][col];
        }
        __syncthreads();
    }

    // Write the result to the output matrix
    int Pidx = Width * BLOCK_SIZE * blockIdx.y + BLOCK_SIZE * blockIdx.x + Width * threadIdx.y + threadIdx.x;
    P[Pidx] = temp;
}