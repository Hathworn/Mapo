#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void testKernel(float *g_idata, float *g_odata)
{
    // Shared memory
    extern __shared__ float sdata[];

    // Access thread id and total number of threads
    const unsigned int tid = threadIdx.x;
    const unsigned int block_size = blockDim.x;
    const unsigned int gid = blockIdx.x * block_size + tid; // Calculate global index

    // Optimized to use global index
    sdata[tid] = g_idata[gid];
    __syncthreads();

    // Perform computations
    sdata[tid] *= block_size;
    __syncthreads();

    // Write data to global memory
    g_odata[gid] = sdata[tid];
}