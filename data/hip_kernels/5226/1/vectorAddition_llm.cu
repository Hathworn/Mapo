#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define tileSize 32

//function for data initialization
void initialization( double *M,  double *N, int arow, int acol, int brow, int bcol);
//(for Debugging) prints out the input data
void printInput( double *M,  double *N, int arow, int acol, int brow,  int bcol);
//(for Debugging) prints out the output data
void printOutput( double *P_C,  double *P_G, int arow, int bcol);

//GPU kernels

__global__ void vectorAddition(const double *A, const double *B, double *C, int numElements)
{
    int gridIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use shared memory for faster access
    __shared__ double cache[tileSize];
    
    // Unrolling loop to improve performance
    for (int i = gridIndex; i < numElements; i += stride * 2)
    {
        double temp = A[i] + B[i];
        C[i] = temp;

        int i2 = i + stride;
        if (i2 < numElements)
        {
            double temp2 = A[i2] + B[i2];
            C[i2] = temp2;
        }
    }
}
```
