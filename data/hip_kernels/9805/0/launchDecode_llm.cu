#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void testPrint() {
    printf("DEVICE PRINT \n");
}

__global__ void launchDecode() {
    // Use shared memory for potential performance improvement
    __shared__ int placeholder; // Placeholder for future shared memory use

    // Use thread index to conditionally print, reduce redundant calls
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("RUNNING\n");
        testPrint();
    }
}