#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ size_t GIDX(size_t row, size_t col, int W) {
    return row * W + col;
}

__global__ void kernel_sub(float* d_f1ptr, float* d_f2ptr, float* d_dt, int H, int W) {

    // Calculate the global thread index
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t col = blockIdx.x * blockDim.x + threadIdx.x;

    // Early return if the thread index is outside the bounds
    if (row < H && col < W) {
        size_t idx = GIDX(row, col, W);
        // Subtract and store result
        d_dt[idx] = d_f2ptr[idx] - d_f1ptr[idx];
    }
}