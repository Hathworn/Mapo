#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define INDEX(row, col, width) ((row) * (width) + (col))

__device__ size_t GIDX(size_t row, size_t col, int H, int W) {
    return INDEX(row, col, W);
}

__global__ void kernel_optflow(float* d_dx1, float* d_dy1, float* d_dx2, float* d_dy2, float* d_dt, float4* uv, float4* uv1, int H, int W) {
    const size_t col = threadIdx.x + blockDim.x * blockIdx.x;
    const size_t row = threadIdx.y + blockDim.y * blockIdx.y;
    const size_t idx = GIDX(row, col, H, W);

    if (row >= H - 2 || row <= 2 || col >= W - 2 || col <= 2) return;
    
    float dx2 = 0.0f, dy2 = 0.0f, dxdy = 0.0f, dxdt = 0.0f, dydt = 0.0f;

    // Precompute index offsets
    int offsets[5] = {-2, -1, 0, 1, 2};

#pragma unroll
    // Loop unrolling and make use of local variables to hold indices
    for (int i = 0; i < 5; i++) {
        int row_offset = row + offsets[i];
        for (int j = 0; j < 5; j++) {
            int col_offset = col + offsets[j];
            int index = INDEX(row_offset, col_offset, W);
            float dx1_val = d_dx1[index];
            float dy1_val = d_dy1[index];
            float dt_val = d_dt[index];

            dx2 += dx1_val * dx1_val;
            dy2 += dy1_val * dy1_val;
            dxdy += dx1_val * dy1_val;
            dxdt += dx1_val * dt_val;
            dydt += dy1_val * dt_val;
        }
    }

    float det = dx2 * dy2 - dxdy * dxdy;
    if (fabsf(det) <= 1.5e-8) {
        uv[idx] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        uv1[idx] = uv[idx];
        return;
    }

    float trace = dx2 + dy2;
    float delta = sqrtf(trace * trace - 4.0f * det);
    if (isnan(delta) || trace - delta <= 0.0002) {
        uv[idx] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
        uv1[idx] = uv[idx];
        return;
    }

    uv[idx].x = (dy2 * -dxdt + dxdy * dydt) / det;
    uv[idx].y = (dxdy * dxdt - dx2 * dydt) / det;
    uv1[idx] = uv[idx];
}