#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "kernel_optflow.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float *d_dx1 = NULL;
hipMalloc(&d_dx1, XSIZE*YSIZE);
float *d_dy1 = NULL;
hipMalloc(&d_dy1, XSIZE*YSIZE);
float *d_dx2 = NULL;
hipMalloc(&d_dx2, XSIZE*YSIZE);
float *d_dy2 = NULL;
hipMalloc(&d_dy2, XSIZE*YSIZE);
float *d_dt = NULL;
hipMalloc(&d_dt, XSIZE*YSIZE);
float4 *uv = NULL;
hipMalloc(&uv, XSIZE*YSIZE);
float4 *uv1 = NULL;
hipMalloc(&uv1, XSIZE*YSIZE);
int H = YSIZE;
int W = XSIZE;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
kernel_optflow<<<gridBlock, threadBlock>>>(d_dx1,d_dy1,d_dx2,d_dy2,d_dt,uv,uv1,H,W);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
kernel_optflow<<<gridBlock, threadBlock>>>(d_dx1,d_dy1,d_dx2,d_dy2,d_dt,uv,uv1,H,W);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
kernel_optflow<<<gridBlock, threadBlock>>>(d_dx1,d_dy1,d_dx2,d_dy2,d_dt,uv,uv1,H,W);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}