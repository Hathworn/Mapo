#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_update_velocities(float4* d_uv, float4* d_velocities_buffer, int numel) {
    // Calculate global thread index
    size_t col = threadIdx.x + blockIdx.x * blockDim.x;
    // Check if within bounds
    if (col >= numel) { return; }
    
    // Update velocities buffer directly without __syncthreads()
    d_velocities_buffer[col] = make_float4(d_uv[col].x, d_uv[col].y, 0, 0);
}