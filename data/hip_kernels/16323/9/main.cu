#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "kernel_convert.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    uchar3 *d_Iin = NULL;
hipMalloc(&d_Iin, XSIZE*YSIZE);
float4 *d_Iout = NULL;
hipMalloc(&d_Iout, XSIZE*YSIZE);
int numel = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
kernel_convert<<<gridBlock, threadBlock>>>(d_Iin,d_Iout,numel);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
kernel_convert<<<gridBlock, threadBlock>>>(d_Iin,d_Iout,numel);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
kernel_convert<<<gridBlock, threadBlock>>>(d_Iin,d_Iout,numel);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}