#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_convert(uchar3* d_Iin, float4* d_Iout, int numel) {
    // Calculate global thread index
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (idx < numel) {
        uchar3 val = d_Iin[idx];
        
        // Convert and store in the output array
        d_Iout[idx] = make_float4(
            val.x * 0.00392156862f, // Pre-compute 1/255.0f for efficiency
            val.y * 0.00392156862f,
            val.z * 0.00392156862f,
            1.0f
        );
    }
}