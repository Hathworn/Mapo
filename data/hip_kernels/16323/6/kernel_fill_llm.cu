#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_fill(float4* d_dx1, float val, int numel) {
    size_t col = threadIdx.x + blockIdx.x * blockDim.x;
    if (col < numel) { // Moved return logic to if condition for clarity
        float4 fillVal = {val, val, val, val}; // Precompute fill value
        d_dx1[col] = fillVal; // Assign fill value directly
    }
}