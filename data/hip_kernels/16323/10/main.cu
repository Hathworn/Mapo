#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "kernel_update.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float4 *d_positions = NULL;
hipMalloc(&d_positions, XSIZE*YSIZE);
float4 *d_og_positions = NULL;
hipMalloc(&d_og_positions, XSIZE*YSIZE);
float4 *d_velocities = NULL;
hipMalloc(&d_velocities, XSIZE*YSIZE);
float *d_masses = NULL;
hipMalloc(&d_masses, XSIZE*YSIZE);
size_t numel = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
kernel_update<<<gridBlock, threadBlock>>>(d_positions,d_og_positions,d_velocities,d_masses,numel);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
kernel_update<<<gridBlock, threadBlock>>>(d_positions,d_og_positions,d_velocities,d_masses,numel);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
kernel_update<<<gridBlock, threadBlock>>>(d_positions,d_og_positions,d_velocities,d_masses,numel);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}