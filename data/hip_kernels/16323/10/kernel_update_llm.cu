#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_update( float4* d_positions, float4* d_og_positions, float4* d_velocities, float* d_masses, size_t numel) {
    // Calculate the global index of the thread
    size_t col = threadIdx.x + blockIdx.x * blockDim.x;
    if (col >= numel) return;

    // Load velocity once to avoid redundant reads
    float4 velocity = d_velocities[col];

    // Use squared magnitude directly for optimization
    float mag = sqrtf(velocity.x * velocity.x + velocity.y * velocity.y) * 0.03f;

    // Use fminf for better performance on float data
    float pos = fminf(mag, 0.50f);

    // Directly assign the new position values
    d_positions[col] = make_float4(
        d_og_positions[col].x,
        d_og_positions[col].y,
        pos,
        0.0f
    );
}