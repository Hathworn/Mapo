#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_grey(float4* d_Iin, float* d_Iout, int numel) {
    size_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Ensure the index is within bounds
    if (idx < numel) {
        float4 pixel = d_Iin[idx];
        
        // Utilize built-in functions for non-redundant computation
        float grey = __fmaf_rn(0.2989f, pixel.x, __fmaf_rn(0.5870f, pixel.y, 0.1140f * pixel.z));
        
        // Write the result to the output array
        d_Iout[idx] = grey;
    }
}