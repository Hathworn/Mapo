#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ size_t GIDX(size_t row, size_t col, int H, int W) {
    return row * W + col;
}

__global__ void kernel_blur(float* d_I, float* d_Ib, int H, int W) {
    // Improved index calculation for boundary check
    size_t row = threadIdx.y + blockDim.y * blockIdx.y;
    size_t col = threadIdx.x + blockDim.x * blockIdx.x;

    // Exit early if out of bounds (reduces redundant checks)
    if (row < KERN_RADIUS || row >= H - KERN_RADIUS || col < KERN_RADIUS || col >= W - KERN_RADIUS) {
        return;
    }

    // Calculate the center index once
    size_t idx = GIDX(row, col, H, W);
    
    // Initialize sum of blurring operation
    float sum = 0.0f;
    int count = 0;
    
    // Use registers to store neighborhood values and accumulate the result
    for (int i = -KERN_RADIUS; i <= KERN_RADIUS; i++) {
        for (int j = -KERN_RADIUS; j <= KERN_RADIUS; j++) {
            sum += d_I[GIDX(row + i, col + j, H, W)] * gaussian_kernel[count];
            count++;
        }
    }
    
    // Write final accumulated value to output
    d_Ib[idx] = sum;
}