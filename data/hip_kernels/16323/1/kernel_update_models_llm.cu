#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_update_models(float4* d_positions, float4* d_modelBuffer, int numel) {
    // Calculate row index using block dimensions for better readability
    size_t idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Early exit if index exceeds number of elements
    if (idx >= numel) return;

    // Directly store the required values in the model buffer
    d_modelBuffer[idx * 4 + 3] = make_float4(
        d_positions[idx].x,
        d_positions[idx].y,
        d_positions[idx].z,
        1.0f  // Use explicit float literal for consistency
    );

    // Remove __syncthreads() as it is not necessary
}