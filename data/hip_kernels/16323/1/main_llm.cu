#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "kernel_update_models_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float4 *d_positions = NULL;
hipMalloc(&d_positions, XSIZE*YSIZE);
float4 *d_modelBuffer = NULL;
hipMalloc(&d_modelBuffer, XSIZE*YSIZE);
int numel = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
kernel_update_models<<<gridBlock, threadBlock>>>(d_positions,d_modelBuffer,numel);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
kernel_update_models<<<gridBlock, threadBlock>>>(d_positions,d_modelBuffer,numel);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
kernel_update_models<<<gridBlock, threadBlock>>>(d_positions,d_modelBuffer,numel);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}