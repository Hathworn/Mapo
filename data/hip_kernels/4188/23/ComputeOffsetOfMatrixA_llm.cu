#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* __restrict__ col_sum, int32_t* __restrict__ output, int32_t N) {
    // Use shared memory for col_sum to reduce global memory access
    extern __shared__ int32_t shared_col_sum[];
    
    // Load col_sum into shared memory, note that this works well for smaller N
    int32_t i = threadIdx.x;
    for (; i < N; i += blockDim.x) {
        shared_col_sum[i] = col_sum[i];
    }
    __syncthreads(); // Ensure all col_sum elements are loaded before computing

    // Compute the output using the shared memory
    i = threadIdx.x;
    for (; i < N; i += blockDim.x) {
        output[blockIdx.x * N + i] = -shared_col_sum[i];
    }
}