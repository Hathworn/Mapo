#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        // Efficiently calculate the output offset using linear indexing.
        *(output + blockIdx.x * N + i) = -row_sum[blockIdx.x];
    }
}