#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Calculate unique index for each thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Iterate over data with stride to ensure all elements are processed
    for (; idx < num_elements; idx += stride) {
        // Assign values to normalized output
        int64_t* dest = to_normalize + idx * 3;  // Pointer arithmetic for direct access
        dest[0] = batch_index;
        dest[1] = class_index;
        dest[2] = static_cast<int64_t>(original[idx]);
    }
}