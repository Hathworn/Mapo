#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Optimize thread indexing: use blockIdx and threadIdx for scalability
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Optimize seed using thread index
    hiprand_init(seed, idx, 0, &state[idx]);
}