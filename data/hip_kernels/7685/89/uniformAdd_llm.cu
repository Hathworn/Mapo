#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    __shared__ float uni;
    
    // Load uniform value into shared memory by the first thread in block
    if (threadIdx.x == 0) {
        uni = uniforms[blockIdx.x + blockOffset];
    }
    
    // Calculate global memory address
    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x;
    
    __syncthreads();
    
    // Use conditional operator directly in index to avoid branching
    g_data[address] += uni;
    if (threadIdx.x + blockDim.x < n) {
        g_data[address + blockDim.x] += uni;
    }
}