#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(const int* A, const int* B, int* C, int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Ensure the current thread does not exceed the size
    if (i < size) {
        int temp = 0;
        // Accumulate temporary sum to reduce global memory accesses
        for(int n = 0; n < 100; n++) {
            temp += A[i] + B[i];
        }
        C[i] += temp;
    }
}