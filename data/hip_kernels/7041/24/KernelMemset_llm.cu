#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int DeviceDefaultStep() {
    return gridDim.x * blockDim.x;
}

__device__ int DeviceDefaultIndex() {
    return blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void KernelMemset(bool *p, int len, bool value) {
    int index = DeviceDefaultIndex();
    int step = DeviceDefaultStep();

    // Optimize loop by reducing branching and allowing more threads to work in parallel
    if (index < len) {
        for (int i = index; i < len; i += step) {
            p[i] = value;
        }
    }
}