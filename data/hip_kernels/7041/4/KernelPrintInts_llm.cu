#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KernelPrintInts(const int* p, int len) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure the index is within bounds
    if (idx < len) {
        printf("%d\n", p[idx]);
    }
}