#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void atemp(double* A, double* y, double* tmp, int NX, int NY)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Early exit if `i` is out of bounds
    if (i >= NY) return;

    double y_sum = 0.0; // Optimize by using a local accumulator
    for (int j = 0; j < NX; j++) {
        y_sum += A[i + j * NY] * tmp[j];
    }
    y[i] += y_sum; // Update global memory once
}