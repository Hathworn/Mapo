#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define FLOAT_N 3214212.01

__global__ void calcmean(double* d_data, double* d_mean, int M, int N)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    // Only threads with valid 'j' proceed
    if (j <= M) {
        double temp_sum = 0.0;
        // Loop unrolling can be applied here for potential further optimization
        for (int i = 1; i <= N; i++) {
            temp_sum += d_data[i * (M + 1) + j];
        }
        d_mean[j] = temp_sum / FLOAT_N;
    }
}