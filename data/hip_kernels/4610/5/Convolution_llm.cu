#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Convolution(double* A, double* B, int I, int J)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Constants declaration outside the loop
    const double c11 = +0.2, c21 = +0.5, c31 = -0.8;
    const double c12 = -0.3, c22 = +0.6, c32 = -0.9;
    const double c13 = +0.4, c23 = +0.7, c33 = +0.1;

    // Optimize boundary and condition checks
    if (i > J && i < I * J - J && (i % J != 0) && ((i + 1) % J != 0)) {
        // Optimize memory access pattern by reducing indirection
        double *Ai = &A[i];
        B[i] = c11 * Ai[-J-1] + c12 * Ai[-1] + c13 * Ai[J-1]
             + c21 * Ai[-J]   + c22 * Ai[0] + c23 * Ai[J]
             + c31 * Ai[-J+1] + c32 * Ai[1] + c33 * Ai[J+1];
    }
}