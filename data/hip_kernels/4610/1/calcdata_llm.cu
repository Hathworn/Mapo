#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define FLOAT_N 3214212.01

__global__ void calcdata(double* d_data, double* d_mean, int M, int N)
{
    // Calculate global index
    int i = blockDim.x * blockIdx.x + threadIdx.x + 1;
    if (i <= N) {  // Limit check to N
        // Use shared memory to optimize memory access
        extern __shared__ double shared_mean[];
        if (threadIdx.x < M) {
            shared_mean[threadIdx.x + 1] = d_mean[threadIdx.x + 1];
        }
        __syncthreads();
        for (int j = 1; j <= M; j++) {
            d_data[i * (M + 1) + j] -= shared_mean[j];
        }
    }
}