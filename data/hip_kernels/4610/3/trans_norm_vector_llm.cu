#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void trans_norm_vector(double* A, double* x, double* y, double* tmp, int NX, int NY)
{
    // Use shared memory to store partial results and reduce global memory access
    extern __shared__ double shared_tmp[];

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int threadId = threadIdx.x;
    
    shared_tmp[threadId] = 0;

    for (int j = 0; j < NY; j++) {
        shared_tmp[threadId] += A[i * NY + j] * x[j];
    }

    __syncthreads();

    // Store result back in global memory
    tmp[i] = shared_tmp[threadId];
}