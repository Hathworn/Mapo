#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global index for the current thread
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure index is within array bounds before performing addition
    if (i < array_size) 
    {
        c[i] = a[i] + b[i];
    }
}