#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setDiagonalToZero(float* d_matrix, uint64_t columnsAndRows)
{
    // Calculate the diagonal index directly
    int matrixIndex = (blockDim.x * blockIdx.x + threadIdx.x) * (columnsAndRows + 1);

    // Check for out of bounds before updating
    if (matrixIndex < columnsAndRows * columnsAndRows)
    {
        // Set diagonal element to zero
        d_matrix[matrixIndex] = 0;
    }
}