#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void normalise(float* result, unsigned int resultLength, float* amps, unsigned int* hits)
{
    // Calculate the absolute thread index
    int absoluteThreadIdx = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure the thread index is within bounds
    if(absoluteThreadIdx >= resultLength) return;

    // Normalize using shared memory for 'hits' to enhance memory access performance
    __shared__ unsigned int sharedHits[256]; // Assumed maximum block size of 256 for illustration
    int localIdx = threadIdx.x / 4;

    if (threadIdx.x % 4 == 0) {
        sharedHits[localIdx] = hits[absoluteThreadIdx / 4];
    }
    __syncthreads();

    // Perform normalization
    result[absoluteThreadIdx] = amps[absoluteThreadIdx] / sharedHits[localIdx];
}