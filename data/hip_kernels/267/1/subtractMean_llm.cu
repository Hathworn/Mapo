#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtractMean(float* d_inputArray, uint64_t inputLength, float d_mean)
{
    // Use local variables for improved memory access
    uint32_t globalThreadIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Eliminate branch divergence by directly returning if condition fails
    if (globalThreadIndex < inputLength)
    {
        // Efficient memory access
        float temp = d_inputArray[globalThreadIndex];
        temp -= d_mean;
        d_inputArray[globalThreadIndex] = temp;
    }
}