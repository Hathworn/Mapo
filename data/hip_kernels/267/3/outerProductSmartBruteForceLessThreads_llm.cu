#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void outerProductSmartBruteForceLessThreads(float* resultMatrix, float* vec, uint64_t vectorLength)
{
    int col = (blockIdx.x * blockDim.x) + threadIdx.x; // Column index
    int row = (blockIdx.y * blockDim.y) + threadIdx.y; // Row index

    // Check bounds
    if (row >= vectorLength || col >= vectorLength)
        return;

    // Optimize thread usage by computing only upper triangular matrix (row <= col)
    if (row > col) {
        int temp = row;
        row = col;
        col = temp;
    }

    int index = (row * vectorLength + col) - (row * (row + 1)) / 2;

    resultMatrix[index] = vec[row] * vec[col];
}