#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void outerProductSmartBruteForce(float* resultMatrix, float* vec, int vectorLength)
{
    // Compute global row and column indices
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Bounds and symmetry check
    if (row < vectorLength && col < vectorLength && row <= col) {
        int index = row * vectorLength + col - (row * (row + 1)) / 2;
        resultMatrix[index] = vec[row] * vec[col]; // Use assignment instead of addition
    }
}