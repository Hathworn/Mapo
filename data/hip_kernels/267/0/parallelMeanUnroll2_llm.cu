#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void parallelMeanUnroll2(float* d_inputArray, uint64_t inputLength, float* d_outputMean)
{
    uint32_t localThreadIndex = threadIdx.x;
    uint32_t globalThreadIndex = blockIdx.x * blockDim.x + localThreadIndex;
    uint32_t sumDataIndex = blockIdx.x * blockDim.x * 2 + localThreadIndex; // Data index for summation

    // Pointer to the block's data
    float* localBlockPointer = d_inputArray + blockIdx.x * blockDim.x * 2;

    // Unrolled addition for better performance, no bounds checking required via shifted index
    if (sumDataIndex + blockDim.x < inputLength)
    {
        localBlockPointer[localThreadIndex] = d_inputArray[sumDataIndex] + d_inputArray[sumDataIndex + blockDim.x];
    }

    // Sync threads within block
    __syncthreads();

    // Optimized in-place reduction using shared pointer, no extra memory access
    for (uint32_t stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (localThreadIndex < stride)
        {
            localBlockPointer[localThreadIndex] += localBlockPointer[localThreadIndex + stride];
        }
        // Sync within reduction loops to ensure data consistency
        __syncthreads();
    }

    // Global mean calculation by first thread only
    if (globalThreadIndex == 0)
    {
        // Accumulating results from all blocks
        float blockSum = 0;
        for (uint32_t i = 0; i < gridDim.x; ++i)
        {
            blockSum += d_inputArray[i * blockDim.x * 2]; // Times 2 for double blocks data
        }

        // Mean computation
        *d_outputMean = blockSum / (inputLength - 1);
    }
}