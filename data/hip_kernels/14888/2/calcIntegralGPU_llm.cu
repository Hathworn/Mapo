#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float f(float x)
{
    return 4.f / (1.f + x * x);
}

__global__ void calcIntegralGPU(float *array, const float h, const long stepCount, const int threads, const int blocks)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce global memory accesses
    __shared__ float shared_data[256]; // Assuming max 256 threads per block
    shared_data[threadIdx.x] = 0.0f;

    for (int i = idx; i < stepCount; i += threads * blocks)
    {
        float x = (i + 0.5f) * h;
        shared_data[threadIdx.x] += f(x);
    }

    // Accumulate shared memory results into global memory
    __syncthreads(); // Ensure all threads in the block have written to shared memory

    // Simple parallel reduction within the block
    if (threadIdx.x == 0) 
    {
        float blockSum = 0.0f;
        for (int i = 0; i < blockDim.x; ++i)
        {
            blockSum += shared_data[i];
        }
        atomicAdd(&array[idx], blockSum); // Using atomic operation to avoid race conditions
    }
}