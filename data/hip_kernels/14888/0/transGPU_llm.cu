#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16

__device__ float f(float x)
{
    return 4.f / (1.f + x * x);
}

__global__ void transGPU(const float *inMatrix, float *outMatrix, const size_t row, const size_t column)
{
    // Use shared memory to reduce global memory accesses
    __shared__ float tile[BLOCK_SIZE][BLOCK_SIZE];

    size_t xIndex = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    size_t yIndex = blockIdx.y * BLOCK_SIZE + threadIdx.y;

    if ((xIndex < column) && (yIndex < row))
    {
        size_t inIndex = yIndex * column + xIndex;
        tile[threadIdx.y][threadIdx.x] = inMatrix[inIndex]; // Loading to shared memory
    }
    __syncthreads(); // Ensure all threads have loaded data into shared memory

    xIndex = blockIdx.y * BLOCK_SIZE + threadIdx.x; // Transpose block offset
    yIndex = blockIdx.x * BLOCK_SIZE + threadIdx.y; // Transpose block offset

    if ((xIndex < row) && (yIndex < column))
    {
        size_t outIndex = yIndex * row + xIndex;
        outMatrix[outIndex] = tile[threadIdx.x][threadIdx.y]; // Store from shared memory to global memory
    }
}