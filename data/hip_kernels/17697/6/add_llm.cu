#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

__global__ void add(const float* __restrict__ x1, const float* __restrict__ x2, float* __restrict__ y, unsigned int len) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure tid is within range to avoid invalid memory access
    if (tid < len) {
        // Perform addition
        y[tid] = x1[tid] + x2[tid];
    }
}