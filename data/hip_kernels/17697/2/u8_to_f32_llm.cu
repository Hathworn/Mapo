#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

__global__ void u8_to_f32(const unsigned char* x, float* y, unsigned int len) {
    const float scale = 1.0f / 255.0f;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll loop for greater efficiency
    for (int i = tid; i < len; i += blockDim.x * gridDim.x) {
        y[i] = scale * x[i];
    }
}