#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

__global__ void cross_entropy_backward(const float* __restrict__ x, float* __restrict__ dx, const float* __restrict__ t, const float* __restrict__ dy, unsigned int len) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure thread is within bounds
    if (tid < len) {
        // Compute gradient
        dx[tid] = dy[0] * (x[tid] - t[tid]);
    }
}