#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void reduce_sum_partial(const float* input, float* output, unsigned int len) {
    // Load a segment of the input vector into shared memory
    __shared__ float partialSum[2 * 256];
    int globalThreadId = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;
    
    // Coalesced global memory reads
    partialSum[t] = ((start + t) < len) ? input[start + t] : 0.0;
    partialSum[blockDim.x + t] = ((start + blockDim.x + t) < len) ? input[start + blockDim.x + t] : 0.0;
    __syncthreads();  // Ensure all loads complete
    
    // Traverse reduction tree with optimized sync
    for (unsigned int stride = blockDim.x; stride > 0; stride /= 2) {
        if (t < stride) {
            partialSum[t] += partialSum[t + stride];
        }
        __syncthreads();  // Ensure all additions complete before next step
    }
    
    // Write the computed sum of the block to the output vector at correct index
    if (t == 0) {
        output[blockIdx.x] = partialSum[0];
    }
}