#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize: Use shared memory and eliminate frequent global memory accesses
__global__ void reverse_conv_filter(const float* __restrict__ x, float beta, float* __restrict__ y, unsigned int filter_len, unsigned int len) {
    extern __shared__ float shared_x[]; // Shared memory for block's portion of x
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    if (tid < len) {
        // Load data to shared memory
        for (int i = local_tid; i < filter_len; i += blockDim.x) {
            shared_x[i] = x[tid * filter_len + i];
        }
        __syncthreads();

        if (beta == 0.0f) {
            for (int i = 0; i < filter_len; ++i) {
                y[tid * filter_len + i] = shared_x[(filter_len - 1) - i]; // Use shared memory
            }
        } else {
            for (int i = 0; i < filter_len; ++i) {
                y[tid * filter_len + i] = shared_x[(filter_len - 1) - i] + beta * y[tid * filter_len + i];
            }
        }
    }
}