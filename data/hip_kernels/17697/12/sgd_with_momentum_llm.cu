#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function for SGD with momentum optimization
__global__ void sgd_with_momentum(float* w, const float* dw, float learning_rate, float momentum, float* v, unsigned int len) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < len) {
        // Update velocity and apply the momentum term
        float velocity = momentum * v[tid] + dw[tid];
        v[tid] = velocity;

        // Update weights with learning rate and current velocity
        w[tid] -= learning_rate * velocity;
    }
}