#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

__global__ void u8_to_one_hot_f32(const unsigned char* x, unsigned int nclasses, float* y, unsigned int len) {
    // Use shared memory to reduce global memory access latency
    extern __shared__ unsigned char x_shared[];
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread copies relevant data to shared memory
    if (tid < len) {
        x_shared[threadIdx.x] = x[tid];
        __syncthreads(); // Synchronize to ensure all data is copied
        
        // Calculate index based on shared memory and set value
        y[tid * nclasses + x_shared[threadIdx.x]] = 1.0f;
    }
}