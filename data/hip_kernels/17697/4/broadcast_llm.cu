#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void broadcast(const float* x, float* y, unsigned int c, unsigned int len) {
    // Calculate global thread ID using grid-stride loop
    for (int tid = blockIdx.x * blockDim.x + threadIdx.x; tid < len; tid += blockDim.x * gridDim.x) {
        // Efficient element assignment
        y[tid] = x[tid % c];
    }
}