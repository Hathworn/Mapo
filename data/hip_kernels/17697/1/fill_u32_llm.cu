#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

__global__ void fill_u32(unsigned int *y, unsigned int elem, unsigned int len) {
    // Calculate absolute thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid-stride loop to allow more threads to handle larger arrays
    for (int index = tid; index < len; index += blockDim.x * gridDim.x) {
        y[index] = elem;
    }
}