#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

__global__ void reduce_sum_final(const float* x, float* y, unsigned int len) {
    // Use shared memory for partial sum
    __shared__ float partial_sum[256];
    
    int tid = threadIdx.x;
    int global_tid = blockIdx.x * blockDim.x + tid;
    
    // Initialize shared memory
    float sum = 0.0f;
    for (int i = global_tid; i < len; i += blockDim.x * gridDim.x) {
        sum += x[i];
    }
    
    // Store each thread's sum in shared memory
    partial_sum[tid] = sum;
    __syncthreads();
    
    // Reduce within block
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            partial_sum[tid] += partial_sum[tid + s];
        }
        __syncthreads();
    }
    
    // Add block's sum to the final result
    if (tid == 0) {
        atomicAdd(y, partial_sum[0]);
    }
}