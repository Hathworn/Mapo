#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

__global__ void broadcast_backward(float* dx, const float* dy, unsigned int c, unsigned int len) {
    // Calculate the global thread index
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterate over the necessary elements using stride
    for (unsigned int i = tid; i < len; i += blockDim.x * gridDim.x) {
        atomicAdd(&dx[i % c], dy[i]); // Efficient atomicAdd operation
    }
}