#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

__global__ void cross_entropy_forward(unsigned int batch_size, unsigned int nclasses, const float* x, const float* t, float* y) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < batch_size) {
        // Load the starting address for the specific batch
        const float* x_batch = x + tid * nclasses;
        const float* t_batch = t + tid * nclasses;
        float* y_batch = y + tid * nclasses;

        // Compute max value of slice
        float m = x_batch[0];
        for(int i = 1; i < nclasses; ++i) {
            m = fmaxf(x_batch[i], m);
        }

        // Subtract max and accumulate softmax
        float s = 0.0f;
        for(int i = 0; i < nclasses; ++i) {
            y_batch[i] = x_batch[i] - m;
            s += expf(y_batch[i]);
        }

        // Compute ln(s)
        float ln_s = logf(s);

        // Update y with the final values
        for(int i = 0; i < nclasses; ++i) {
            y_batch[i] = (ln_s - y_batch[i]) * t_batch[i];
        }
    }
}