#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod, const size_t lod_len, size_t* out_lod0) {
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure thread index is within the bounds
    if (index < lod_len) {
        // Optimized memory access with temporary variable
        size_t lod_value = in_lod[index];
        out_lod0[index] = lod_value - num_erased[lod_value];
    }
}