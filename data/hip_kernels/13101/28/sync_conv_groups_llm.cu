#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() {
    // Optimized kernel to use a barrier for synchronization
    __syncthreads();  // Ensure threads within the block synchronize
}