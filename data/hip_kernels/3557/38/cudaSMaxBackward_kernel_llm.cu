#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSMaxBackward_kernel(unsigned int size, float* diffInput, const unsigned int idx, unsigned int* argMax, const float beta, float* result)
{
    // Calculate the global thread index
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the total number of threads used
    const unsigned int stride = blockDim.x * gridDim.x;

    // Loop over elements with strided access pattern
    for (unsigned int i = index; i < size; i += stride) {
        // Use ternary operator to update result based on argMax match
        result[i] = (argMax[i] == idx) ? (diffInput[i] + beta * result[i]) : beta * result[i];
    }
}