#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDinv_kernel(unsigned int size, const double *x, double *y)
{
    // Calculate global index and stride for the loop
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    if (index < size) { // Check index bounds before entering loop
        for (unsigned int i = index; i < size; i += stride) {
            y[i] = 1.0 / x[i]; // Perform division
        }
    }
}