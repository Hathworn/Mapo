#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSMult_kernel(unsigned int size, float* a, float* b, const float beta, float* result)
{
    // Calculate global thread index
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory for caching
    extern __shared__ float shared[];

    if (beta != 0.0f) {
        for (unsigned int i = index; i < size; i += stride) {
            // Load data into shared memory
            shared[threadIdx.x] = a[i] * b[i];
            __syncthreads(); // Synchronize threads within the block
            // Write result back to global memory
            result[i] = shared[threadIdx.x] + beta * result[i];
        }
    } else {
        for (unsigned int i = index; i < size; i += stride) {
            // Load data into shared memory
            shared[threadIdx.x] = a[i] * b[i];
            __syncthreads(); // Synchronize threads within the block
            // Write result back to global memory
            result[i] = shared[threadIdx.x];
        }
    }
}