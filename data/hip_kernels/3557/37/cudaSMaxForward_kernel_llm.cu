#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSMaxForward_kernel(unsigned int size, float* input, float* maxVal, const unsigned int idx, unsigned int* argMax)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory to reduce global memory accesses
    __shared__ float sharedMax;
    __shared__ unsigned int sharedIdx;

    if(threadIdx.x == 0) {
        sharedMax = -FLT_MAX; // Initialize with smallest possible float value
        sharedIdx = 0;
    }
    __syncthreads();

    float localMax = sharedMax;
    unsigned int localIdx = sharedIdx;

    for (unsigned int i = index; i < size; i += stride) {
        if (input[i] > localMax) {
            localMax = input[i];
            localIdx = idx;
        }
    }

    if(threadIdx.x == 0) {
        sharedMax = localMax;
        sharedIdx = localIdx;
    }
    __syncthreads();

    for (unsigned int i = index; i < size; i += stride) {
        if (sharedMax > maxVal[i]) {
            maxVal[i] = sharedMax;
            argMax[i] = sharedIdx;
        }
    }
}