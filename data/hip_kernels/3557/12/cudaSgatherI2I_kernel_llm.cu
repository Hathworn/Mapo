#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSgatherI2I_kernel(const int* keys, const int* indicesX, const int* indicesY, const int* indicesK, int* outX, int* outY, int* outK, unsigned int nbElements)
{
    // Calculate unique global thread index
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we do not read/write out of bounds
    if (index < nbElements) {
        const int key = keys[index];
        outX[index] = indicesX[key];
        outY[index] = indicesY[key];
        outK[index] = indicesK[key];
    }
}