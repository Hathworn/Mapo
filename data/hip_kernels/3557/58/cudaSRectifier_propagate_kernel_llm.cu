#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSRectifier_propagate_kernel(float* x, float* y, unsigned int size, float leakSlope, float clipping)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Optimization: loop unrolling for better performance
    #pragma unroll 4
    for (unsigned int i = index; i < size; i += stride) {
        float value = x[i];

        // Utilize conditional operator for more concise code and potential speed improvements
        y[i] = (value > 0.0f) ? ((clipping > 0.0f) ? min(value, clipping) : value) : leakSlope * value;
    }
}