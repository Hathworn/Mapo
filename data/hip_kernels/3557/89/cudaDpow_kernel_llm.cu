#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaDpow_kernel(unsigned int size, double power, const double *x, double *y)
{
    // Optimize memory access by ensuring threads don't exceed array bounds
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    if(index < size) {
        y[index] = pow(x[index], power); // Use 'pow' for double precision
    }

    for (unsigned int i = index + stride; i < size; i += stride) {
        y[i] = pow(x[i], power); // Use 'pow' for double precision
    }
}