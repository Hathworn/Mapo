#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSPadding_kernel(unsigned int nbOutputs, unsigned int outputWidth, unsigned int outputHeight, unsigned int nbChannels, unsigned int batchSize, unsigned int inputWidth, unsigned int inputHeight, int leftPad, int rightPad, int topPad, int botPad, const float* input, float* outputs)
{
    // Calculate input and output offsets
    const unsigned int inputOffset = (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels * inputWidth * inputHeight;
    const unsigned int outputOffset = (blockIdx.z * blockDim.z + threadIdx.z) * nbOutputs * outputWidth * outputHeight;

    // Calculate the number of channels to process
    const unsigned int nbCh = min(nbChannels, nbOutputs);

    // Loop over channels with blockIdx.x as index
    for (unsigned int ch = blockIdx.x; ch < nbCh; ch += gridDim.x) {
        // Loop over output height with threadIdx.y as index
        for (unsigned int oy = threadIdx.y; oy < outputHeight; oy += blockDim.y) {
            // Loop over output width with threadIdx.x as index
            for (unsigned int ox = threadIdx.x; ox < outputWidth; ox += blockDim.x) {
                float outputValue = 0.0f;
                int ix = static_cast<int>(ox) - leftPad;
                int iy = static_cast<int>(oy) - topPad;

                // Check if indices are within input bounds
                if (ix >= 0 && ix < static_cast<int>(inputWidth) && iy >= 0 && iy < static_cast<int>(inputHeight)) {
                    outputValue = input[ix + iy * inputWidth + ch * inputWidth * inputHeight + inputOffset];
                }
                
                // Write output value
                outputs[ox + oy * outputWidth + ch * outputWidth * outputHeight + outputOffset] = outputValue;
            }
        }
    }
}