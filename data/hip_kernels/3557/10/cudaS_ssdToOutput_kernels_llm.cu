#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaS_ssdToOutput_kernels(
    unsigned int batchSize, unsigned int nbClass, unsigned int nbAnchors, unsigned int channelWidth,
    unsigned int channelHeight, unsigned int nbProposals, unsigned int* nbValidROIs, unsigned int cls,
    unsigned int totalParts, unsigned int totalTemplates, unsigned int maxParts, unsigned int maxTemplates,
    unsigned int cumulParts, unsigned int cumulTemplates, unsigned int nbParts, unsigned int nbTemplates,
    float xRatio, float yRatio, float xOutputRatio, float yOutputRatio, const float* roi_bbox,
    const float* roi_anchors, const float* anchors, const float* inputs_parts, const float* inputs_templates, float* outputs)
{
    const int batchPos = blockIdx.z;
    const int proposal = (threadIdx.x & 0x1f) + blockIdx.x * blockDim.x;
    const int ptIdx = blockIdx.y;

    const int nbDetectedObject = static_cast<int>(nbValidROIs[batchPos]);
    const int nbIdx = 6;

    if (proposal < nbProposals)
    {
        const unsigned int n = proposal + cls * nbProposals + batchPos * nbProposals * nbClass;

        if (proposal < nbDetectedObject)
        {
            if (ptIdx == 0)
            {
                // Copy bbox data
                for (int i = 0; i < 5; ++i)
                {
                    outputs[i + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] =
                        roi_bbox[i + 5 * proposal + batchPos * nbProposals * 5];
                }
                outputs[5 + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = static_cast<float>(cls);
            }

            if (ptIdx < nbParts && nbParts > 0)
            {
                // Calculate part prediction
                const unsigned int xa = roi_anchors[0 + 5 * proposal + batchPos * nbProposals * 5];
                const unsigned int ya = roi_anchors[1 + 5 * proposal + batchPos * nbProposals * 5];
                const unsigned int k = roi_anchors[2 + 5 * proposal + batchPos * nbProposals * 5];

                const int offset = xa + ya * channelWidth + (k * nbParts * 2 + cumulParts + ptIdx * 2) * channelHeight * channelWidth;
                const int partIdx = offset + batchPos * channelHeight * channelWidth * nbAnchors * 2 * totalParts;
                const float partY = inputs_parts[partIdx];
                const float partX = inputs_parts[partIdx + 1];

                // Calculate anchor properties
                const int xa0 = static_cast<int>(anchors[cls * nbAnchors * 4 + k * 4] + xa * xRatio);
                const int ya0 = static_cast<int>(anchors[cls * nbAnchors * 4 + k * 4 + 1] + ya * yRatio);
                const int xa1 = static_cast<int>(anchors[cls * nbAnchors * 4 + k * 4 + 2] + xa * xRatio);
                const int ya1 = static_cast<int>(anchors[cls * nbAnchors * 4 + k * 4 + 3] + ya * yRatio);

                const int wa = xa1 - xa0;
                const int ha = ya1 - ya0;
                const float xac = xa0 + wa / 2.0;
                const float yac = ya0 + ha / 2.0;
                const float predPartY = (partY * ha + yac) * yOutputRatio;
                const float predPartX = (partX * wa + xac) * xOutputRatio;

                outputs[ptIdx * 2 + nbIdx + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = predPartY;
                outputs[ptIdx * 2 + nbIdx + 1 + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = predPartX;
            }
            else if (ptIdx < maxParts && maxParts > 0)
            {
                // Fill remaining parts with zeros
                outputs[ptIdx * 2 + nbIdx + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = 0.0f;
                outputs[ptIdx * 2 + nbIdx + 1 + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = 0.0f;
            }
            
            if (ptIdx < nbTemplates && nbTemplates > 0)
            {
                // Calculate template prediction
                const unsigned int xa = roi_anchors[0 + 5 * proposal + batchPos * nbProposals * 5];
                const unsigned int ya = roi_anchors[1 + 5 * proposal + batchPos * nbProposals * 5];
                const unsigned int k = roi_anchors[2 + 5 * proposal + batchPos * nbProposals * 5];

                const int offset = xa + ya * channelWidth + (k * nbTemplates * 3 + cumulTemplates + ptIdx * 3) * channelHeight * channelWidth;
                const int templateIdx = offset + batchPos * channelHeight * channelWidth * nbAnchors * 3 * totalTemplates;
                
                const float templateY = expf(inputs_templates[templateIdx]);
                const float templateX = expf(inputs_templates[templateIdx + 1]);
                const float templateZ = expf(inputs_templates[templateIdx + 2]);

                outputs[ptIdx * 3 + maxParts * 2 + nbIdx + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = templateY;
                outputs[ptIdx * 3 + maxParts * 2 + nbIdx + 1 + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = templateX;
                outputs[ptIdx * 3 + maxParts * 2 + nbIdx + 2 + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = templateZ;
            }
            else if (ptIdx < maxTemplates && maxTemplates > 0)
            {
                // Fill remaining templates with zeros
                outputs[ptIdx * 3 + maxParts * 2 + nbIdx + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = 0.0f;
                outputs[ptIdx * 3 + maxParts * 2 + nbIdx + 1 + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = 0.0f;
                outputs[ptIdx * 3 + maxParts * 2 + nbIdx + 2 + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = 0.0f;
            }
        }
        else
        {
            // Fill the output with zeros for invalid proposals
            for (int i = 0; i < nbIdx; ++i)
            {
                outputs[i + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = 0.0f;
            }

            if (ptIdx < maxParts && maxParts > 0)
            {
                outputs[ptIdx * 2 + nbIdx + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = 0.0f;
                outputs[ptIdx * 2 + nbIdx + 1 + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = 0.0f;
            }

            if (ptIdx < maxTemplates && maxTemplates > 0)
            {
                outputs[ptIdx * 3 + maxParts * 2 + nbIdx + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = 0.0f;
                outputs[ptIdx * 3 + maxParts * 2 + nbIdx + 1 + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = 0.0f;
                outputs[ptIdx * 3 + maxParts * 2 + nbIdx + 2 + n * (nbIdx + maxParts * 2 + maxTemplates * 3)] = 0.0f;
            }
        }
    }
}