#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaUpdateMetrics_kernel(float *inputs, int *activity, long long unsigned int *firingRate, long long unsigned int *totalFiringRate, long long int *outputsActivity, long long int *totalOutputsActivity, unsigned int inputsDimX, unsigned int inputsDimY, unsigned int inputsDimZ)
{
    const unsigned int inputSize = inputsDimZ * inputsDimX * inputsDimY;
    const unsigned int batchInputOffset = blockIdx.z * inputSize;
    
    // Using shared memory and launch optimally within dimensions
    extern __shared__ int sharedMem[];

    unsigned int channel = blockIdx.x;
    unsigned int y = threadIdx.y;
    unsigned int x = threadIdx.x;

    if (channel < inputsDimZ && y < inputsDimY && x < inputsDimX) {
        const unsigned int inputsIdx = x + y * inputsDimX + channel * inputsDimX * inputsDimY;

        // Load data to shared memory
        sharedMem[threadIdx.y * blockDim.x + threadIdx.x] = round(inputs[inputsIdx + batchInputOffset]);
        __syncthreads();

        int value = sharedMem[threadIdx.y * blockDim.x + threadIdx.x];
        unsigned int event = value == 0 ? 0 : 1;

        activity[inputsIdx + batchInputOffset] = event;
        firingRate[inputsIdx + batchInputOffset] += event;
        totalFiringRate[inputsIdx + batchInputOffset] += event;
        outputsActivity[inputsIdx + batchInputOffset] += value;
        totalOutputsActivity[inputsIdx + batchInputOffset] += value;
    }
}