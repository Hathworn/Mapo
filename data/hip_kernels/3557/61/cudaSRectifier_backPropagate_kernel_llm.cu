#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSRectifier_backPropagate_kernel(float* x, float* dx, unsigned int size, float leakSlope, float clipping)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        // Precompute conditionals outside loop to avoid branch divergence
        float multiplier;
        if (clipping > 0.0f) {
            multiplier = (x[i] > clipping) ? 0.0f : ((x[i] > 0.0f) ? 1.0f : leakSlope);
        } else {
            multiplier = (x[i] > 0.0f) ? 1.0f : leakSlope;
        }
        dx[i] *= multiplier; // Apply precomputed multiplier
    }
}