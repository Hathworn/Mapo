#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSEuclideanSumBackward_kernel(unsigned int size, float* diffInput, float* input, float* output, const float scale, const float beta, float* result)
{
    // Calculate one-dimensional global thread index
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory to preload the beta
    __shared__ float sharedBeta;
    if (threadIdx.x == 0) {
        sharedBeta = beta;
    }
    __syncthreads();

    // Combine the two loops with a conditional statement
    for (unsigned int i = index; i < size; i += stride) {
        float outputVal = output[i];
        float computedValue = (outputVal != 0.0f) ? diffInput[i] * scale * (input[i] / outputVal) : 0.0f;
        result[i] = (sharedBeta != 0.0f) ? computedValue + sharedBeta * result[i] : computedValue;
    }
}