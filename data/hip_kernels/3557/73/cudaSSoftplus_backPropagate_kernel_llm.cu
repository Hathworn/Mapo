#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSSoftplus_backPropagate_kernel(float* x, float* dx, unsigned int size)
{
    // Use shared memory for frequently accessed data
    __shared__ float shared_x[256]; // Assuming 256 threads per block as an example. Adjust size if needed.
    
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    for (unsigned int i = index; i < size; i += stride) {
        // Load data to shared memory
        shared_x[threadIdx.x] = x[i];

        __syncthreads(); // Ensure all threads have loaded their data

        // Use shared memory for repeated computation
        dx[i] *= (1.0f - exp(-shared_x[threadIdx.x]));

        __syncthreads(); // Ensure all threads have completed computation before next iteration
    }
}