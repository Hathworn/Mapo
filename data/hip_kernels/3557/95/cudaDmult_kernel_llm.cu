#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDmult_kernel(unsigned int size, const double *x1, const double *x2, double *y)
{
    // Get the global index using blockIdx and threadIdx
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for faster access
    __shared__ double shared_x1[1024];
    __shared__ double shared_x2[1024];
    
    // Load data into shared memory if within bounds
    if (index < size) {
        shared_x1[threadIdx.x] = x1[index];
        shared_x2[threadIdx.x] = x2[index];
    }

    __syncthreads();

    // Perform multiplication using shared memory data
    if (index < size) {
        y[index] = shared_x1[threadIdx.x] * shared_x2[threadIdx.x];
    }
}