#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSGatherRP_kernel(unsigned int inputSizeX, unsigned int inputSizeY, unsigned int nbAnchors, unsigned int batchSize, const float* inputs, const int* i, const int* j, const int* k, const int* b, const int* mask, float* outputs, int* anchors, unsigned int topN, const unsigned int nbProposals)
{
    const int batchPos = blockIdx.z;
    const int sortOffset = batchPos * topN;
    const int index = (threadIdx.x & 0x1f) + blockIdx.x * blockDim.x;
    const int totalIndex = index + sortOffset;
    const int batchIndex = index + batchPos * nbProposals;

    if (index < nbProposals) {
        // Precompute common expressions for reuse
        int maskVal = mask[totalIndex + sortOffset];

        unsigned int commonOffset1 = maskVal * inputSizeX;
        unsigned int commonOffset2 = maskVal * inputSizeX * inputSizeY;

        unsigned int baseOffset = b[maskVal] * nbAnchors * commonOffset2 * 6;

        unsigned int xIdx = commonOffset1 + j[maskVal] * inputSizeX + (k[maskVal] + nbAnchors) * commonOffset2 + baseOffset;
        unsigned int yIdx = commonOffset1 + j[maskVal] * inputSizeX + (k[maskVal] + 2 * nbAnchors) * commonOffset2 + baseOffset;
        unsigned int wIdx = commonOffset1 + j[maskVal] * inputSizeX + (k[maskVal] + 3 * nbAnchors) * commonOffset2 + baseOffset;
        unsigned int hIdx = commonOffset1 + j[maskVal] * inputSizeX + (k[maskVal] + 4 * nbAnchors) * commonOffset2 + baseOffset;

        anchors[0 + (batchIndex) * 4] = i[maskVal];
        anchors[1 + (batchIndex) * 4] = j[maskVal];
        anchors[2 + (batchIndex) * 4] = k[maskVal];
        anchors[3 + (batchIndex) * 4] = b[maskVal];

        outputs[0 + (batchIndex) * 4] = inputs[xIdx];
        outputs[1 + (batchIndex) * 4] = inputs[yIdx];
        outputs[2 + (batchIndex) * 4] = inputs[wIdx];
        outputs[3 + (batchIndex) * 4] = inputs[hIdx];
    }
}