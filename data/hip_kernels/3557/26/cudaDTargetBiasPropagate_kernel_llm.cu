#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDTargetBiasPropagate_kernel(unsigned int size, const double bias, const double* inputs, const double* diffInputs, double* outputs)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory for faster access
    __shared__ double sharedInputs[BLOCK_SIZE];
    __shared__ double sharedDiffInputs[BLOCK_SIZE];

    // Load inputs and diffInputs into shared memory
    if (index < size) {
        sharedInputs[threadIdx.x] = inputs[index];
        sharedDiffInputs[threadIdx.x] = diffInputs[index];
    }
    __syncthreads();

    // Ensure the computation only proceeds for valid index
    if (index < size) {
        outputs[index] = sharedInputs[threadIdx.x];
        if (sharedDiffInputs[threadIdx.x] > 0.0 && sharedInputs[threadIdx.x] > -bias) {
            outputs[index] += bias;
        }
    }
}