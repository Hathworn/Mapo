#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaUZeroInit_kernel(unsigned int size, unsigned int* data)
{
    // Use shared memory for faster access
    extern __shared__ unsigned int shared_data[];

    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        shared_data[threadIdx.x] = 0U; // Store in shared memory
        data[i] = shared_data[threadIdx.x]; // Copy from shared memory to global memory
    }
}