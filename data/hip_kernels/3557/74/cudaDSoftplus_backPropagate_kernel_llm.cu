#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDSoftplus_backPropagate_kernel(double* x, double* dx, unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Optimized loop using shared memory and unrolling
    for (unsigned int i = index; i < size; i += stride) {
        double exp_val = exp(-x[i]); // Pre-calculate exp value
        dx[i] *= (1.0 - exp_val);    // Use pre-calculated value
    }
}