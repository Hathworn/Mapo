#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSGatherRP_kernel(unsigned int inputSizeX, unsigned int inputSizeY, unsigned int nbAnchors, unsigned int batchSize,
                                     const float* inputs, const float* i, const float* j, const float* k, const float* b, const int* mask,
                                     float* outputs, const unsigned int topN, const unsigned int nbProposals)
{
    const int batchPos = blockIdx.z;
    const int sortOffset = batchPos * topN;
    int index = (threadIdx.x & 0x1f) + blockIdx.x * blockDim.x;
    const int totalIndex = index + sortOffset;
    const int batchIndex = index + batchPos * nbProposals;

    if (index < nbProposals)
    {
        // Compute indices for each dimension using reused mask offset
        unsigned int maskOffset = mask[totalIndex] + sortOffset;
        unsigned int anchorOffset = nbAnchors * inputSizeX * inputSizeY;

        unsigned int xIdx = i[maskOffset] + j[maskOffset] * inputSizeX + (k[maskOffset] + nbAnchors) * anchorOffset + b[maskOffset] * nbAnchors * anchorOffset * 6;
        unsigned int yIdx = i[maskOffset] + j[maskOffset] * inputSizeX + (k[maskOffset] + 2 * nbAnchors) * anchorOffset + b[maskOffset] * nbAnchors * anchorOffset * 6;
        unsigned int wIdx = i[maskOffset] + j[maskOffset] * inputSizeX + (k[maskOffset] + 3 * nbAnchors) * anchorOffset + b[maskOffset] * nbAnchors * anchorOffset * 6;
        unsigned int hIdx = i[maskOffset] + j[maskOffset] * inputSizeX + (k[maskOffset] + 4 * nbAnchors) * anchorOffset + b[maskOffset] * nbAnchors * anchorOffset * 6;

        // Store results in outputs using offsets in a single operation per proposal
        outputs[0 + (batchIndex) * 4] = inputs[xIdx];
        outputs[1 + (batchIndex) * 4] = inputs[yIdx];
        outputs[2 + (batchIndex) * 4] = inputs[wIdx];
        outputs[3 + (batchIndex) * 4] = inputs[hIdx];
    }
}