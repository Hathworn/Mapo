#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSReduceIndex_kernel(const unsigned int inputSize, const unsigned int inputBatchOffset, const unsigned int outputBatchOffset, const float* valueThreshold, const float* inputs, int* outputMap, float* scores)
{
    // Compute global index using block and thread index
    const int batchPos = blockIdx.z;
    const int clsPos = blockIdx.y;
    const int index = (threadIdx.x & 0x1f) + blockIdx.x * blockDim.x;

    // Calculate input and output indices
    const int inputIndex = index + inputSize * clsPos + batchPos * inputBatchOffset;
    const int outputIndex = index + inputSize * clsPos + batchPos * outputBatchOffset;

    // Check if index is within input size
    if (index < inputSize)
    {
        float value = inputs[inputIndex];

        // Use ternary operator to simplify conditional assignments
        outputMap[outputIndex] = (value >= valueThreshold[clsPos]) ? index : -1;
        scores[outputIndex] = (value >= valueThreshold[clsPos]) ? value : -1.0f;
    }
}