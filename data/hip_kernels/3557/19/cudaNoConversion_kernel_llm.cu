#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaNoConversion_kernel(float * data, float * tickData, float * tickActivity, float scaling, unsigned int inputDimX, unsigned int inputDimY, unsigned int inputDimZ)
{
    const unsigned int inputSize = inputDimX * inputDimY * inputDimZ;
    const unsigned int batchOffset = blockIdx.x * inputSize;

    // Optimize memory access by using shared memory for frequently accessed data
    __shared__ float sharedData[BLOCK_SIZE]; // Assume BLOCK_SIZE is defined and suitable

    unsigned int idx = threadIdx.x;
    while (idx < inputSize) {
        sharedData[threadIdx.x] = data[idx + batchOffset]; // Load data into shared memory

        __syncthreads(); // Ensure all threads have loaded their data

        float value = sharedData[threadIdx.x];
        tickData[idx + batchOffset] = scaling * value;
        tickActivity[idx + batchOffset] += scaling * value;

        idx += blockDim.x;
    }
}