#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDSoftplus_propagate_kernel(double* x, double* y, unsigned int size)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory to reduce global memory access; assuming blockDim.x <= size
    __shared__ double s_data[1024]; // Adjust size as needed based on blockDim.x

    for (unsigned int i = index; i < size; i += stride) {
        // Load data into shared memory
        s_data[threadIdx.x] = x[i];
        __syncthreads();

        // Compute softplus and write to output; use shared memory data
        y[i] = log(1.0 + exp(s_data[threadIdx.x]));
        __syncthreads();
    }
}