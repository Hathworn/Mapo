#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSNearestNeighborKernel(const float* input, size_t inputSizeX, size_t inputSizeY, float* output, size_t outputSizeX, size_t outputSizeY, size_t nbChannels, size_t batchSize)
{
    // Calculate input and output offsets
    const size_t inputOffset = blockIdx.z * nbChannels * inputSizeY * inputSizeX + threadIdx.z * nbChannels * inputSizeY * inputSizeX;
    const size_t outputOffset = blockIdx.z * nbChannels * outputSizeY * outputSizeX + threadIdx.z * nbChannels * outputSizeY * outputSizeX;

    // Precompute multipliers for coordinate mapping
    const float multy = static_cast<float>(inputSizeY) / static_cast<float>(outputSizeY);
    const float multx = static_cast<float>(inputSizeX) / static_cast<float>(outputSizeX);

    // Calculate channel index
    size_t channel = blockIdx.x * blockDim.z + threadIdx.z;
    while (channel < nbChannels) {
        // Calculate destination pixel position
        size_t oy = threadIdx.y + blockIdx.y * blockDim.y;
        while (oy < outputSizeY) {
            size_t ox = threadIdx.x + blockIdx.x * blockDim.x;
            while (ox < outputSizeX) {
                // Calculate source pixel position
                const size_t iy = static_cast<size_t>(oy * multy);
                const size_t ix = static_cast<size_t>(ox * multx);

                // Perform the nearest neighbor mapping
                output[outputOffset + channel * outputSizeY * outputSizeX + oy * outputSizeX + ox] =
                    input[inputOffset + channel * inputSizeY * inputSizeX + iy * inputSizeX + ix];

                ox += blockDim.x * gridDim.x; // Advance in x-direction
            }
            oy += blockDim.y * gridDim.y; // Advance in y-direction
        }
        channel += blockDim.z * gridDim.z; // Advance channel-wise
    }
}