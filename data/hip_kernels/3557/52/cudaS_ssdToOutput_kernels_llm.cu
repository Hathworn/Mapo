#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaS_ssdToOutput_kernels(
    unsigned int batchSize, unsigned int nbClass, unsigned int nbAnchors, unsigned int channelWidth, unsigned int channelHeight, unsigned int nbProposals, unsigned int* nbValidROIs, unsigned int cls, unsigned int totalParts, unsigned int totalTemplates, unsigned int maxParts, unsigned int maxTemplates, unsigned int cumulParts, unsigned int cumulTemplates, unsigned int nbParts, unsigned int nbTemplates, float xRatio, float yRatio, float xOutputRatio, float yOutputRatio, const float* roi_bbox, const float* roi_anchors, const float* anchors, const float* inputs_parts, const float* inputs_templates, float* outputs)
{
    const int batchPos = blockIdx.z;
    const int proposal = (threadIdx.x & 0x1f) + blockIdx.x*blockDim.x;
    const int ptIdx = blockIdx.y;
    const unsigned int nbAnchorPerCls = nbAnchors;
    const int nbDetectedObject  = (int) nbValidROIs[batchPos];
    const int nbIdx = 6;

    if(proposal < nbProposals)
    {
        const unsigned int n = proposal + cls*nbProposals + batchPos*nbProposals*nbClass;

        if(proposal < nbDetectedObject)
        {
            // Efficient initialization of the output
            float* outBasePtr = &outputs[n*(nbIdx + maxParts*2 + maxTemplates*3)];
            const float* roiBasePtr = &roi_bbox[5*proposal + batchPos*nbProposals*5];

            if(ptIdx == 0)
            {
                outBasePtr[0] = roiBasePtr[0];
                outBasePtr[1] = roiBasePtr[1];
                outBasePtr[2] = roiBasePtr[2];
                outBasePtr[3] = roiBasePtr[3];
                outBasePtr[4] = roiBasePtr[4];
                outBasePtr[5] = (float) cls;
            }

            if(ptIdx < nbParts && totalParts > 0)
            {
                const unsigned int xa = roiBasePtr[0];
                const unsigned int ya = roiBasePtr[1];
                const unsigned int k  = roiBasePtr[2];

                const int yIdx = xa + ya * channelWidth + (k * nbParts * 2 + cumulParts + ptIdx * 2) * channelHeight * channelWidth + batchPos * channelHeight * channelWidth * nbAnchorPerCls * 2 * totalParts;
                const int xIdx = xa + ya * channelWidth + (k * nbParts * 2 + cumulParts + ptIdx * 2 + 1) * channelHeight * channelWidth + batchPos * channelHeight * channelWidth * nbAnchorPerCls * 2 * totalParts;

                const float partY = inputs_parts[yIdx];
                const float partX = inputs_parts[xIdx];

                const int xa0 = (int)(anchors[cls * 4 * nbAnchorPerCls + k * 4] + xa * xRatio);
                const int ya0 = (int)(anchors[cls * 4 * nbAnchorPerCls + k * 4 + 1] + ya * yRatio);
                const int xa1 = (int)(anchors[cls * 4 * nbAnchorPerCls + k * 4 + 2] + xa * xRatio);
                const int ya1 = (int)(anchors[cls * 4 * nbAnchorPerCls + k * 4 + 3] + ya * yRatio);

                const int wa = xa1 - xa0;
                const int ha = ya1 - ya0;

                const float xac = xa0 + wa / 2.0;
                const float yac = ya0 + ha / 2.0;

                outBasePtr[ptIdx * 2 + 0 + nbIdx] = ((partY) * ha + yac) * yOutputRatio;
                outBasePtr[ptIdx * 2 + 1 + nbIdx] = ((partX) * wa + xac) * xOutputRatio;
            }
            else if(ptIdx < maxParts && totalParts > 0)
            {
                outBasePtr[ptIdx * 2 + 0 + nbIdx] = 0.0;
                outBasePtr[ptIdx * 2 + 1 + nbIdx] = 0.0;
            }

            if(ptIdx < nbTemplates && totalTemplates > 0)
            {
                const unsigned int xa = roiBasePtr[0];
                const unsigned int ya = roiBasePtr[1];
                const unsigned int k  = roiBasePtr[2];

                const int yIdx = xa + ya * channelWidth + (k * nbTemplates * 3 + cumulTemplates + ptIdx * 3) * channelHeight * channelWidth + batchPos * channelHeight * channelWidth * nbAnchorPerCls * 3 * totalTemplates;
                const int xIdx = xa + ya * channelWidth + (k * nbTemplates * 3 + cumulTemplates + ptIdx * 3 + 1) * channelHeight * channelWidth + batchPos * channelHeight * channelWidth * nbAnchorPerCls * 3 * totalTemplates;
                const int zIdx = xa + ya * channelWidth + (k * nbTemplates * 3 + cumulTemplates + ptIdx * 3 + 2) * channelHeight * channelWidth + batchPos * channelHeight * channelWidth * nbAnchorPerCls * 3 * totalTemplates;

                outBasePtr[ptIdx * 3 + maxParts * 2 + 0 + nbIdx] = expf(inputs_templates[yIdx]);
                outBasePtr[ptIdx * 3 + maxParts * 2 + 1 + nbIdx] = expf(inputs_templates[xIdx]);
                outBasePtr[ptIdx * 3 + maxParts * 2 + 2 + nbIdx] = expf(inputs_templates[zIdx]);
            }
            else if(ptIdx < maxTemplates && totalTemplates > 0)
            {
                outBasePtr[ptIdx * 3 + maxParts * 2 + 0 + nbIdx] = 0.0;
                outBasePtr[ptIdx * 3 + maxParts * 2 + 1 + nbIdx] = 0.0;
                outBasePtr[ptIdx * 3 + maxParts * 2 + 2 + nbIdx] = 0.0;
            }

        }
        else
        {
            float* outBasePtr = &outputs[n*(nbIdx + maxParts*2 + maxTemplates*3)];
            outBasePtr[0] = 0.0;
            outBasePtr[1] = 0.0;
            outBasePtr[2] = 0.0;
            outBasePtr[3] = 0.0;
            outBasePtr[4] = 0.0;

            if(ptIdx < maxParts && totalParts > 0)
            {
                outBasePtr[ptIdx * 2 + 0 + nbIdx] = 0.0;
                outBasePtr[ptIdx * 2 + 1 + nbIdx] = 0.0;
            }

            if(ptIdx < maxTemplates && totalTemplates > 0)
            {
                outBasePtr[ptIdx * 3 + maxParts * 2 + 0 + nbIdx] = 0.0;
                outBasePtr[ptIdx * 3 + maxParts * 2 + 1 + nbIdx] = 0.0;
                outBasePtr[ptIdx * 3 + maxParts * 2 + 2 + nbIdx] = 0.0;
            }
        }
    }
}