#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSZeroInit_kernel(unsigned int size, float* data)
{
    // Calculate global index
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride to handle larger datasets efficiently
    unsigned int stride = blockDim.x * gridDim.x;

    // Efficient initialization using stride looping
    for (; index < size; index += stride)
    {
        data[index] = 0.0f;
    }
}