#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDSaturation_backPropagate_kernel(double* x, double* dx, unsigned int size, double threshold)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    if (threshold != 0.0) {
        for (unsigned int i = index; i < size; i += stride) {
            // Use ternary operation directly to avoid branching
            dx[i] *= (x[i] > -threshold && x[i] < threshold) ? 1.0 : 0.0;
        }
    }
}