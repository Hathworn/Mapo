#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSReduceIndex_kernel(const unsigned int inputSize, const unsigned int inputBatchOffset, const unsigned int outputBatchOffset, const unsigned int channelsWidth, const unsigned int channelsHeight, const unsigned int nbAnchors, const float* valueThreshold, const float* inputs, int* outputMap, float* scores)
{
    const int batchPos = blockIdx.z;
    const int clsPos = blockIdx.y;

    // Calculate the index using shared memory for faster access
    __shared__ float sharedValueThreshold;
    if (threadIdx.x == 0) {
        sharedValueThreshold = valueThreshold[clsPos];
    }
    __syncthreads();

    const int index = (threadIdx.x & 0x1f) + blockIdx.x * blockDim.x;
    const int inputIndex = index + inputSize * clsPos + batchPos * inputBatchOffset;
    const int outputIndex = index + inputSize * clsPos + batchPos * outputBatchOffset;

    if (index < inputSize) {
        float value = inputs[inputIndex];

        // Use ternary operator for concise conditional assignment
        int outputMapValue = (value >= sharedValueThreshold) ? index : -255;
        float scoreValue = (value >= sharedValueThreshold) ? value : -255.0;

        outputMap[outputIndex] = outputMapValue;
        scores[outputIndex] = scoreValue;
    }
}