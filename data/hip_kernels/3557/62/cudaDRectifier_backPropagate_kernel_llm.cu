#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDRectifier_backPropagate_kernel(double* x, double* dx, unsigned int size, double leakSlope, double clipping)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory for faster access if beneficial
    extern __shared__ double s_x[];
  
    for (unsigned int i = index; i < size; i += stride) {
        // Load elements into shared memory
        s_x[threadIdx.x] = x[i];
        __syncthreads();

        double xi = s_x[threadIdx.x];
        if (clipping > 0.0) {
            dx[i] *= (xi > clipping) ? 0.0 : (xi > 0.0) ? 1.0 : leakSlope;
        } else {
            dx[i] *= (xi > 0.0) ? 1.0 : leakSlope;
        }
    }
}