#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSSaturation_propagate_kernel(float* x, float* y, unsigned int size, float threshold)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    if (threshold == 0.0f) return; // Early exit if threshold is zero

    for (unsigned int i = index; i < size; i += stride) {
        float value = x[i];
        // Use fminf and fmaxf for more efficient clamping
        y[i] = fmaxf(fminf(value, threshold), -threshold);
    }
}