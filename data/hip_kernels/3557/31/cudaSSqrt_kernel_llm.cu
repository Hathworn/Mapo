#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSSqrt_kernel(unsigned int size, float* data)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    if (index < size)  // Ensure valid index before processing
    {
        data[index] = sqrt(data[index]);
        for (unsigned int i = index + stride; i < size; i += stride)
        {
            data[i] = sqrt(data[i]);  // Process remaining elements within bounds
        }
    }
}