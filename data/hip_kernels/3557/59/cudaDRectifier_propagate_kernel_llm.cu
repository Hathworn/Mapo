#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDRectifier_propagate_kernel(double* x, double* y, unsigned int size, double leakSlope, double clipping)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Loop optimization with unrolling
    for (unsigned int i = index; i < size; i += stride * 4) {
        double value1 = x[i];
        double value2 = (i + stride < size) ? x[i + stride] : 0.0;  // Bounds check
        double value3 = (i + stride * 2 < size) ? x[i + stride * 2] : 0.0;  // Bounds check
        double value4 = (i + stride * 3 < size) ? x[i + stride * 3] : 0.0;  // Bounds check

        if (clipping > 0.0) {
            y[i] = (value1 > 0.0) ? min(value1, clipping) : leakSlope * value1;
            if (i + stride < size)
                y[i + stride] = (value2 > 0.0) ? min(value2, clipping) : leakSlope * value2;
            if (i + stride * 2 < size)
                y[i + stride * 2] = (value3 > 0.0) ? min(value3, clipping) : leakSlope * value3;
            if (i + stride * 3 < size)
                y[i + stride * 3] = (value4 > 0.0) ? min(value4, clipping) : leakSlope * value4;
        } else {
            y[i] = (value1 > 0.0) ? value1 : leakSlope * value1;
            if (i + stride < size)
                y[i + stride] = (value2 > 0.0) ? value2 : leakSlope * value2;
            if (i + stride * 2 < size)
                y[i + stride * 2] = (value3 > 0.0) ? value3 : leakSlope * value3;
            if (i + stride * 3 < size)
                y[i + stride * 3] = (value4 > 0.0) ? value4 : leakSlope * value4;
        }
    }
}