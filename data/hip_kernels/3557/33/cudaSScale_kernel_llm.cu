#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSScale_kernel(unsigned int size, float* input, const float scale, const float shift, const float beta, float* result)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory if applicable and unroll loop if needed (max 8)
    if (beta != 0.0f) {
        #pragma unroll
        for (unsigned int i = index; i < size; i += stride) {
            result[i] = input[i] * scale + shift + beta * result[i];
        }
    }
    else {
        #pragma unroll
        for (unsigned int i = index; i < size; i += stride) {
            result[i] = input[i] * scale + shift;
        }
    }
}