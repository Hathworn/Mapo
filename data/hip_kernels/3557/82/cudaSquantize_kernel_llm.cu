#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSquantize_kernel(float* x, float* y, unsigned int size, float minVal, float maxVal, unsigned int quantizationLevels, bool truncate)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    if (quantizationLevels > 1) {
        const float scaling = (maxVal - minVal) / (float)(quantizationLevels - 1);

        for (unsigned int i = index; i < size; i += stride) {
            // Clamp value within minVal and maxVal
            const float clamped = fminf(fmaxf(x[i], minVal), maxVal);

            // Perform quantization with or without rounding
            y[i] = (truncate ? (int)((clamped - minVal) / scaling) : (int)roundf((clamped - minVal) / scaling))
                   * scaling + minVal;
        }
    }
    else {
        for (unsigned int i = index; i < size; i += stride) {
            // Assign binary values based on sign
            y[i] = (x[i] >= 0.0f) ? 1.0f : -1.0f;
        }
    }
}