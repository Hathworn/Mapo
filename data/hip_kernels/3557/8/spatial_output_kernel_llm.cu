#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void spatial_output_kernel(unsigned int nbClass, unsigned int targetHeight, unsigned int targetWidth, float threshold, float* targetData, uint32_t* outputEstimated)
{
    const int batchInputOffset = targetWidth * targetHeight * nbClass * blockIdx.z;
    const int batchOutputOffset = targetWidth * targetHeight * blockIdx.z;

    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < targetWidth * targetHeight; i += stride)
    {
        unsigned int outputMax = 0;

        if (nbClass > 1)
        {
            // Initialize maxVal and outputMax with the first class
            float maxVal = targetData[i + batchInputOffset];

            // Loop unrolling might help with smaller values of nbClass
            for (unsigned int cls = 1; cls < nbClass; ++cls) {
                const float tmp = targetData[i + cls * targetWidth * targetHeight + batchInputOffset];
                if (tmp > maxVal) {
                    outputMax = cls;
                    maxVal = tmp;
                }
            }
            outputEstimated[i + batchOutputOffset] = outputMax;
        }
        else if(nbClass == 1)
        {
            // Directly compute estimatedLabel
            const int estimatedLabel = (targetData[i + batchInputOffset] > threshold);
            outputEstimated[i + batchOutputOffset] = estimatedLabel;
        }
    }
}