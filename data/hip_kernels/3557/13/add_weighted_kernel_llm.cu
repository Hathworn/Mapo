#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_weighted_kernel(unsigned int batchSize, unsigned int nbOutputs, unsigned int outputsHeight, unsigned int outputsWidth, float* estimated_labels, unsigned int nbChannels, unsigned int image_height, unsigned int image_width, float* input_image, unsigned char* workspace, float alpha)
{
    const int batchEstimatedOffset = nbOutputs * outputsHeight * outputsWidth * blockIdx.z;
    const int batchImageOffset = nbChannels * image_height * image_width * blockIdx.z;

    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < outputsWidth * outputsHeight; i += stride) {
        unsigned int outputMax = 0;
        float maxVal = estimated_labels[i + batchEstimatedOffset];

        // Loop unrolling and branch reduction for better warp efficiency
        for (unsigned int cls = 1; cls < nbOutputs; cls += 4) {
            float tmp1 = (cls < nbOutputs) ? estimated_labels[i + cls * outputsWidth * outputsHeight + batchEstimatedOffset] : -INFINITY;
            float tmp2 = (cls + 1 < nbOutputs) ? estimated_labels[i + (cls + 1) * outputsWidth * outputsHeight + batchEstimatedOffset] : -INFINITY;
            float tmp3 = (cls + 2 < nbOutputs) ? estimated_labels[i + (cls + 2) * outputsWidth * outputsHeight + batchEstimatedOffset] : -INFINITY;
            float tmp4 = (cls + 3 < nbOutputs) ? estimated_labels[i + (cls + 3) * outputsWidth * outputsHeight + batchEstimatedOffset] : -INFINITY;

            if (tmp1 > maxVal) { outputMax = cls; maxVal = tmp1; }
            if (tmp2 > maxVal) { outputMax = cls + 1; maxVal = tmp2; }
            if (tmp3 > maxVal) { outputMax = cls + 2; maxVal = tmp3; }
            if (tmp4 > maxVal) { outputMax = cls + 3; maxVal = tmp4; }
        }

        // Prefetching color and input_image values to reduce repeated memory accesses
        float color0 = colors[outputMax % 4][0] * alpha;
        float color1 = colors[outputMax % 4][1] * alpha;
        float color2 = colors[outputMax % 4][2] * alpha;

        unsigned int imgIdxBase = i + batchImageOffset;
        unsigned int imgOffset = image_height * image_width;
        float imgVal0 = input_image[imgIdxBase];
        float imgVal1 = input_image[imgIdxBase + imgOffset];
        float imgVal2 = input_image[imgIdxBase + 2 * imgOffset];

        unsigned char ch0 = (unsigned char)max(color0, min(255.0, color0 + imgVal0));
        unsigned char ch1 = (unsigned char)max(color1, min(255.0, color1 + imgVal1));
        unsigned char ch2 = (unsigned char)max(color2, min(255.0, color2 + imgVal2));

        unsigned int workIdxBase = i * 3 + batchImageOffset;
        workspace[workIdxBase] = ch0;
        workspace[workIdxBase + 1] = ch1;
        workspace[workIdxBase + 2] = ch2;
    }
}