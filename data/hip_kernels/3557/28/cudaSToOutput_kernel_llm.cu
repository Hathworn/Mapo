#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSToOutput_kernel(const unsigned int nbProposals, const unsigned int scoreIdx, const unsigned int nbCls, 
                                     const unsigned int nbOutputs, const unsigned int maxParts, const unsigned int maxTemplates, 
                                     bool generateParts, bool generateTemplates, const int* numPartsPerClass, const int* numTemplatesPerClass, 
                                     const int* maxCls, const float* ROIEst, const int* predictionIndex, const float* partsPrediction, 
                                     const float* partsVisibilityPrediction, const float* templatesPrediction, float* outputs) {
    const int batchPos = blockIdx.z * nbProposals;
    const int index = threadIdx.x + blockIdx.x * blockDim.x;  // Use full threadIdx for index calculation

    if (index < nbProposals) {
        const unsigned int inputIdx = index * 4 * (nbCls - scoreIdx) + batchPos * 4 * (nbCls - scoreIdx);
        unsigned int outputIdx = 0;
        unsigned offset = 0;

        // Simplify conditionals
        if (nbOutputs == 4) {
            outputIdx = index * 4 + batchPos * 4;
        } else if (nbOutputs == 5) {
            outputIdx = index * 5 + batchPos * 5;
        } else if (generateParts && generateTemplates) {
            outputIdx = (index + batchPos) * (5 + maxParts * 3 + maxTemplates * 3);
        } else if (generateTemplates) {
            outputIdx = (index + batchPos) * (5 + maxTemplates * 3);
        } else if (generateParts) {
            outputIdx = (index + batchPos) * (5 + maxParts * 3);
        }

        // Output ROIEst data
        for (int i = 0; i < 4; ++i) {
            outputs[i + outputIdx] = ROIEst[i + inputIdx];
        }

        offset += 4;

        if (nbOutputs > 4) {
            int cls = maxCls[index + batchPos];
            outputs[4 + outputIdx] = cls > -1 ? (float)cls : 0.0f;
            offset += 1;
        }

        if (generateParts) {
            const int predProp = predictionIndex[(index + batchPos) * 2 + 0];
            const int predCls = predictionIndex[(index + batchPos) * 2 + 1];

            if (predCls > -1) {
                // Iterate over parts
                for (unsigned int part = 0; part < numPartsPerClass[predCls]; ++part) {
                    const unsigned int partIdx = batchPos * maxParts * 2 * nbCls + 
                                                 predProp * maxParts * 2 * nbCls +
                                                 predCls * maxParts * 2 + 
                                                 part * 2;
                    outputs[offset + part * 2 + outputIdx] = partsPrediction[partIdx];
                    outputs[offset + part * 2 + 1 + outputIdx] = partsPrediction[partIdx + 1];
                }
                // Zero-out remaining parts
                for (int idx = numPartsPerClass[predCls]; idx < maxParts; ++idx) {
                    outputs[offset + idx * 2 + outputIdx] = 0.0f;
                    outputs[offset + idx * 2 + 1 + outputIdx] = 0.0f;
                }
            }
            offset += 2 * maxParts;

            if (predCls > -1) {
                // Parts visibility
                for (unsigned int part = 0; part < numPartsPerClass[predCls]; ++part) {
                    const unsigned int partVisibilityIdx = batchPos * maxParts * nbCls +
                                                           predProp * maxParts * nbCls +
                                                           predCls * maxParts +
                                                           part;
                    outputs[offset + part + outputIdx] = partsVisibilityPrediction[partVisibilityIdx];
                }
                // Fill remaining visibility with -1
                for (int idx = numPartsPerClass[predCls]; idx < maxParts; ++idx) {
                    outputs[offset + idx + outputIdx] = -1.0f;
                }
            }
            offset += maxParts;
        }

        if (generateTemplates) {
            const int predProp = predictionIndex[(index + batchPos) * 2 + 0];
            const int predCls = predictionIndex[(index + batchPos) * 2 + 1];

            if (predCls > -1) {
                // Iterate over templates
                for (unsigned int tpl = 0; tpl < numTemplatesPerClass[predCls]; ++tpl) {
                    unsigned int templateIdx = batchPos * maxTemplates * 3 * nbCls +
                                               predProp * maxTemplates * 3 * nbCls +
                                               predCls * maxTemplates * 3 +
                                               tpl * 3;
                    for (int i = 0; i < 3; ++i) {
                        outputs[offset + tpl * 3 + i + outputIdx] = templatesPrediction[i + templateIdx];
                    }
                }
                // Zero-out remaining templates
                for (int idx = numTemplatesPerClass[predCls]; idx < maxTemplates; ++idx) {
                    for (int i = 0; i < 3; ++i) {
                        outputs[offset + numTemplatesPerClass[predCls] * 3 + idx * 3 + i + outputIdx] = 0.0f;
                    }
                }
            }
        }
    }
}
```
