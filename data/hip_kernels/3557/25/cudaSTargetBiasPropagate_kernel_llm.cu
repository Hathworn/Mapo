#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSTargetBiasPropagate_kernel(unsigned int size, const float bias, const float* inputs, const float* diffInputs, float* outputs)
{
    // Calculating the global index for current thread
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop with stride to cover all elements
    for (unsigned int i = index; i < size; i += gridDim.x * blockDim.x) {
        float inputValue = inputs[i]; // Load input value
        float diffInputValue = diffInputs[i]; // Load diffInput value
        
        // Apply bias conditionally
        outputs[i] = inputValue + ((diffInputValue > 0.0f && inputValue > -bias) ? bias : 0.0f);
    }
}