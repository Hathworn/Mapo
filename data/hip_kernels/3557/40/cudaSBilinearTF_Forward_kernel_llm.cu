#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSBilinearTF_Forward_kernel( unsigned int outputWidth, unsigned int outputHeight, unsigned int nbChannels, unsigned int batchSize, unsigned int inputWidth, unsigned int inputHeight, const unsigned int* yLowIdx, const unsigned int* yHighIdx, const float* yInter, const unsigned int* xLowIdx, const unsigned int* xHighIdx, const float* xInter, const float* input, float* outputs)
{
    const unsigned int inputOffset = (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels * inputWidth * inputHeight;
    const unsigned int outputOffset = (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels * outputWidth * outputHeight;

    // Using shared memory for input indices and interpolation values
    __shared__ unsigned int shared_yLowIdx[1024];
    __shared__ unsigned int shared_yHighIdx[1024];
    __shared__ float shared_yInter[1024];
    __shared__ unsigned int shared_xLowIdx[1024];
    __shared__ unsigned int shared_xHighIdx[1024];
    __shared__ float shared_xInter[1024];

    // Load shared memory
    shared_yLowIdx[threadIdx.x] = yLowIdx[threadIdx.x];
    shared_yHighIdx[threadIdx.x] = yHighIdx[threadIdx.x];
    shared_yInter[threadIdx.x] = yInter[threadIdx.x];
    shared_xLowIdx[threadIdx.x] = xLowIdx[threadIdx.x];
    shared_xHighIdx[threadIdx.x] = xHighIdx[threadIdx.x];
    shared_xInter[threadIdx.x] = xInter[threadIdx.x];

    __syncthreads(); // Synchronize threads after loading shared memory

    for (unsigned int ch = blockIdx.x; ch < nbChannels; ch += gridDim.x) {
        for (unsigned int oy = threadIdx.y; oy < outputHeight; oy += blockDim.y) {
            for (unsigned int ox = threadIdx.x; ox < outputWidth; ox += blockDim.x) {
                const unsigned int indexTL = shared_xLowIdx[ox] + shared_yLowIdx[oy] * inputWidth + ch * inputWidth * inputHeight + inputOffset;
                const unsigned int indexTR = shared_xHighIdx[ox] + shared_yLowIdx[oy] * inputWidth + ch * inputWidth * inputHeight + inputOffset;
                const unsigned int indexBL = shared_xLowIdx[ox] + shared_yHighIdx[oy] * inputWidth + ch * inputWidth * inputHeight + inputOffset;
                const unsigned int indexBR = shared_xHighIdx[ox] + shared_yHighIdx[oy] * inputWidth + ch * inputWidth * inputHeight + inputOffset;

                const float top_left = input[indexTL];
                const float top_right = input[indexTR];
                const float bottom_left = input[indexBL];
                const float bottom_right = input[indexBR];

                const float top = top_left + (top_right - top_left) * shared_xInter[ox];
                const float bottom = bottom_left + (bottom_right - bottom_left) * shared_xInter[ox];

                outputs[ox + oy * outputWidth + ch * outputWidth * outputHeight + outputOffset] = top + (bottom - top) * shared_yInter[oy];
            }
        }
    }
}