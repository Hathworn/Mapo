#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSmult_kernel(unsigned int size, const float *x1, const float *x2, float *y)
{
    // Use shared memory for buffering input, improves memory access patterns
    extern __shared__ float sdata[];

    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        // Load data into shared memory
        sdata[threadIdx.x] = x1[i] * x2[i];

        __syncthreads(); // Synchronize threads within a block

        // Write the result from shared memory back to global memory
        y[i] = sdata[threadIdx.x];
    }
}