#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDclamp_kernel(double* __restrict__ x, unsigned int size, double minVal, double maxVal)
{
    // Calculate index and stride for accessing elements
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Process elements using stride
    for (unsigned int i = index; i < size; i += stride) {
        // Clamp value within minVal and maxVal
        double val = x[i];
        x[i] = (val < minVal) ? minVal : (val > maxVal) ? maxVal : val;
    }
}