#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSToOutput_kernel( unsigned int nbProposals, const unsigned int scoreIdx, const unsigned int nbCls, const unsigned int nbOutputs, const unsigned int maxParts, const unsigned int maxTemplates, bool generateParts, bool generateTemplates, const unsigned int* numPartsPerClass, const unsigned int* numTemplatesPerClass, const int* maxCls, const float* inputs, const int* predictionIndex, const float* partsPrediction, const float* partsVisibilityPrediction, const float* templatesPrediction, float* outputs)
{
    const int batchPos = blockIdx.z*nbProposals;
    const int index = (threadIdx.x & 0x1f) + blockIdx.x*blockDim.x;

    if(index < nbProposals)
    {
        // Precompute common indices to reduce redundancy
        const unsigned int inputIdxBase = index * 4 * (nbCls - scoreIdx) + batchPos * 4 * (nbCls - scoreIdx);
        unsigned int outputIdx = 0;
        unsigned int offset = 4; 
        
        // Streamline computation for outputIdx based on conditions 
        if (nbOutputs == 4) {
            outputIdx = index * 4 + batchPos * 4;
        }
        else if (nbOutputs == 5) {
            outputIdx = index * 5 + batchPos * 5;
        }
        else {
            outputIdx = (index + batchPos) * (5 + (generateParts ? maxParts*3 : 0) + (generateTemplates ? maxTemplates*3 : 0));
        }

        // Store common output data
        for(int i = 0; i < 4; ++i) {
            outputs[i + outputIdx] = inputs[i + inputIdxBase];
        }

        if (nbOutputs > 4) {
            int cls = maxCls[index + batchPos];
            outputs[4 + outputIdx] = cls > -1 ? static_cast<float>(cls) : 0.0;
            offset += 1;
        }

        if (generateParts) {
            const int predProp = predictionIndex[(index + batchPos) * 2];
            const int predCls = predictionIndex[(index + batchPos) * 2 + 1];

            if (predCls > -1) {
                unsigned int partBase = batchPos * maxParts * 2 * nbCls + predProp * maxParts * 2 * nbCls + predCls * maxParts * 2;
                for (unsigned int part = 0; part < numPartsPerClass[predCls]; ++part) {
                    for (int j = 0; j < 2; ++j) {
                        outputs[j + offset + part * 2 + outputIdx] = partsPrediction[j + partBase + part * 2];
                    }
                }
                for (int idx = numPartsPerClass[predCls]; idx < maxParts; ++idx) {
                    for (int j = 0; j < 2; ++j) {
                        outputs[j + offset + numPartsPerClass[predCls] * 2 + idx * 2 + outputIdx] = 0.0;
                    }
                }
            }
            offset += maxParts * 2;

            if (predCls > -1) {
                unsigned int partVisibilityBase = batchPos * maxParts * nbCls + predProp * maxParts * nbCls + predCls * maxParts;
                for (unsigned int part = 0; part < numPartsPerClass[predCls]; ++part) {
                    outputs[offset + part + outputIdx] = partsVisibilityPrediction[partVisibilityBase + part];
                }
                for (int idx = numPartsPerClass[predCls]; idx < maxParts; ++idx) {
                    outputs[offset + numPartsPerClass[predCls] + idx + outputIdx] = -1.0;
                }
            }
            offset += maxParts;
        }

        if (generateTemplates) {
            const int predProp = predictionIndex[(index + batchPos) * 2];
            const int predCls = predictionIndex[(index + batchPos) * 2 + 1];

            if (predCls > -1) {
                unsigned int templateBase = batchPos * maxTemplates * 3 * nbCls + predProp * maxTemplates * 3 * nbCls + predCls * maxTemplates * 3;
                for (unsigned int tpl = 0; tpl < numTemplatesPerClass[predCls]; ++tpl) {
                    for (int j = 0; j < 3; ++j) {
                        outputs[j + offset + tpl * 3 + outputIdx] = templatesPrediction[j + templateBase + tpl * 3];
                    }
                }
                for (int idx = numTemplatesPerClass[predCls]; idx < maxTemplates; ++idx) {
                    for (int j = 0; j < 3; ++j) {
                        outputs[j + offset + numTemplatesPerClass[predCls] * 3 + idx * 3 + outputIdx] = 0.0;
                    }
                }
            }
        }
    }
}