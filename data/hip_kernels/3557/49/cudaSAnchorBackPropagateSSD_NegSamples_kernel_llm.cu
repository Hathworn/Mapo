#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSAnchorBackPropagateSSD_NegSamples_kernel(const float* inputCls, float* diffOutputsCls, const int* keySamples, const int nbSamples, const int nbPositive, const unsigned int nbAnchors, const unsigned int outputsHeight, const unsigned int outputsWidth, const unsigned int batchSize)
{
    // Calculate global index with warp size alignment
    int index = (threadIdx.x + blockIdx.x * blockDim.x);

    if (index < nbSamples) {
        // Read index only once
        int indexSamples = keySamples[index];
        
        // Compute error and update only once
        float error = inputCls[indexSamples];
        diffOutputsCls[indexSamples] = -error / nbPositive;
    }
}