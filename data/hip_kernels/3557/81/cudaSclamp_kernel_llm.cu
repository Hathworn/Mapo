#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSclamp_kernel(float* x, unsigned int size, float minVal, float maxVal)
{
    // Calculate global thread index
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop using stride based on grid and block dimensions
    for (unsigned int i = index; i < size; i += blockDim.x * gridDim.x) {
        // Apply clamp operation
        float val = x[i];
        x[i] = min(max(val, minVal), maxVal);
    }
}