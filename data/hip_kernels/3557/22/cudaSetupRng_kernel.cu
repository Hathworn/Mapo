#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSetupRng_kernel(hiprandState * state, unsigned int seed)
{
int id = threadIdx.x + blockIdx.x * blockDim.x;
// Each thread gets the same seed but a different sequence number
hiprand_init(seed, id, 0, &state[id]);
}