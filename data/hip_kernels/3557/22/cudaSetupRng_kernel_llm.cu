#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSetupRng_kernel(hiprandState *state, unsigned int seed)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use blockStride loop to handle more threads than available
    int totalThreads = gridDim.x * blockDim.x;
    for (int i = id; i < totalThreads; i += totalThreads)
    {
        // Initialize the RNG state for each thread
        hiprand_init(seed, i, 0, &state[i]);
    }
}