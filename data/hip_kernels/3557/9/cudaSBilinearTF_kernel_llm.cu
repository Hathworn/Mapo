#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSBilinearTF_kernel(unsigned int outputWidth, unsigned int outputHeight, unsigned int nbChannels, unsigned int batchSize, unsigned int inputWidth, unsigned int inputHeight, const unsigned int* yLowIdx, const unsigned int* yHighIdx, const float* yInter, const unsigned int* xLowIdx, const unsigned int* xHighIdx, const float* xInter, const float* input, float* outputs)
{
    const unsigned int inputOffset = ((blockIdx.z * blockDim.z) + threadIdx.z) * nbChannels * inputWidth * inputHeight;
    const unsigned int outputOffset = ((blockIdx.z * blockDim.z) + threadIdx.z) * nbChannels * outputWidth * outputHeight;

    // Loop fusion and unrolling to improve memory coalescing and reduce loop overhead
    for (unsigned int ch = blockIdx.x; ch < nbChannels; ch += gridDim.x) {
        for (unsigned int oy = threadIdx.y; oy < outputHeight; oy += blockDim.y) {
            for (unsigned int ox = threadIdx.x; ox < outputWidth; ox += blockDim.x) {
                const unsigned int xLow = xLowIdx[ox];
                const unsigned int xHigh = xHighIdx[ox];
                const unsigned int yLow = yLowIdx[oy];
                const unsigned int yHigh = yHighIdx[oy];
                const float xInt = xInter[ox];
                const float yInt = yInter[oy];
                const unsigned int chOffset = ch * inputWidth * inputHeight;
                
                const unsigned int indexTL = xLow + (yLow * inputWidth) + chOffset + inputOffset;
                const unsigned int indexTR = xHigh + (yLow * inputWidth) + chOffset + inputOffset;
                const unsigned int indexBL = xLow + (yHigh * inputWidth) + chOffset + inputOffset;
                const unsigned int indexBR = xHigh + (yHigh * inputWidth) + chOffset + inputOffset;

                const float top_left = input[indexTL];
                const float top_right = input[indexTR];
                const float bottom_left = input[indexBL];
                const float bottom_right = input[indexBR];

                const float top = top_left + (top_right - top_left) * xInt;
                const float bottom = bottom_left + (bottom_right - bottom_left) * xInt;

                outputs[ox + (oy * outputWidth) + (ch * outputWidth * outputHeight) + outputOffset] = top + (bottom - top) * yInt;
            }
        }
    }
}