#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaDquantize_kernel(double* x, double* y, unsigned int size, double minVal, double maxVal, unsigned int quantizationLevels, bool truncate)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    if (quantizationLevels > 1) {
        const double scaling = (maxVal - minVal) / (double)(quantizationLevels - 1);

        // Optimize by reducing redundant condition checks
        for (unsigned int i = index; i < size; i += stride) {
            double clamped = x[i];
            if (clamped < minVal) clamped = minVal;
            else if (clamped > maxVal) clamped = maxVal;

            double quantizedValue = (clamped - minVal) / scaling;
            if (truncate)
                quantizedValue = (int)quantizedValue;
            else
                quantizedValue = (int)round(quantizedValue);

            y[i] = quantizedValue * scaling + minVal;
        }
    } else {
        for (unsigned int i = index; i < size; i += stride)
            y[i] = ((x[i] >= 0.0) ? 1.0 : -1.0);
    }
}