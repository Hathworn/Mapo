#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSBilinearTF_BackWard_kernel(
    unsigned int outputWidth, unsigned int outputHeight,
    unsigned int nbChannels, unsigned int batchSize, 
    unsigned int inputWidth, unsigned int inputHeight, 
    const float scaleX, const float scaleY,
    const float* diffInput, float* diffOutputs)
{
    const unsigned int inputOffset = 
        (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels * inputWidth * inputHeight;
    
    const unsigned int outputOffset = 
        (blockIdx.z * blockDim.z + threadIdx.z) * nbChannels * outputWidth * outputHeight;
    
    unsigned int ch = blockIdx.x;
    while (ch < nbChannels) {  // Loop unrolling for channels
        unsigned int oy = threadIdx.y;
        while (oy < outputHeight) {  // Loop unrolling for outputHeight
            const float in_y = oy * scaleY;
            const int top_y_index = (int)(floorf(in_y));
            const int bottom_y_index = (in_y < inputHeight - 1) ? ceilf(in_y) : inputHeight - 1;
            
            const float y_lerp = in_y - top_y_index;
            const float inverse_y_lerp = 1.0f - y_lerp;
            
            unsigned int ox = threadIdx.x;
            while (ox < outputWidth) {  // Loop unrolling for outputWidth
                const float in_x = ox * scaleX;
                const int left_x_index = (int)(floorf(in_x));
                const int right_x_index = (in_x < inputWidth - 1) ? ceilf(in_x) : inputWidth - 1;
                
                const float x_lerp = in_x - left_x_index;
                const float inverse_x_lerp = 1.0f - x_lerp;
                
                const unsigned int inLeftTopIdx = 
                    left_x_index + top_y_index * inputWidth + ch * inputWidth * inputHeight + inputOffset;
                const unsigned int inRightTopIdx = 
                    right_x_index + top_y_index * inputWidth + ch * inputWidth * inputHeight + inputOffset;
                const unsigned int inLeftBotIdx = 
                    left_x_index + bottom_y_index * inputWidth + ch * inputWidth * inputHeight + inputOffset;
                const unsigned int inRightBotIdx = 
                    right_x_index + bottom_y_index * inputWidth + ch * inputWidth * inputHeight + inputOffset;
                
                const unsigned int outIdx = 
                    ox + oy * outputWidth + ch * outputWidth * outputHeight + outputOffset;
                const float outData = diffInput[outIdx];
                
                atomicAdd(&diffOutputs[inLeftTopIdx], outData * inverse_y_lerp * inverse_x_lerp);  // Use atomicAdd
                atomicAdd(&diffOutputs[inRightTopIdx], outData * inverse_y_lerp * x_lerp);
                atomicAdd(&diffOutputs[inLeftBotIdx], outData * y_lerp * inverse_x_lerp);
                atomicAdd(&diffOutputs[inRightBotIdx], outData * y_lerp * x_lerp);
                
                ox += blockDim.x;
            }
            oy += blockDim.y;
        }
        ch += gridDim.x;
    }
}