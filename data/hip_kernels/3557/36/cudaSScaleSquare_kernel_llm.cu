#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSScaleSquare_kernel(unsigned int size, float* input, const float scale, const float shift, const float beta, float* result)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Use shared memory to avoid redundant access
    __shared__ float s_result[256];
    
    float val = (index < size) ? input[index] * input[index] * scale + shift : 0.0f;
    if (beta != 0.0f) {
        for (unsigned int i = index; i < size; i += stride) {
            val = input[i] * input[i] * scale + shift + beta * result[i];
            s_result[threadIdx.x] = val; // Store value in shared memory
            result[i] = s_result[threadIdx.x];
        }
    }
    else {
        for (unsigned int i = index; i < size; i += stride) {
            s_result[threadIdx.x] = val; // Store value in shared memory
            result[i] = s_result[threadIdx.x];
        }
    }
}