#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaSScaleSign_kernel(unsigned int size, float* input, float* sign, const float scale, const float beta, float* result)
{
    // Use shared memory to reduce redundant access to global memory
    extern __shared__ float shared_sign[];

    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    const unsigned int tid = threadIdx.x;

    // Load data into shared memory
    for (unsigned int i = tid; i < size; i += blockDim.x) {
        shared_sign[i] = sign[i];
    }
    __syncthreads();

    if (beta != 0.0f) {
        for (unsigned int i = index; i < size; i += stride) {
            const float sgn = (shared_sign[i] >= 0) ? 1.0f : -1.0f;
            result[i] = input[i] * sgn * scale + beta * result[i];
        }
    }
    else {
        for (unsigned int i = index; i < size; i += stride) {
            const float sgn = (shared_sign[i] >= 0) ? 1.0f : -1.0f;
            result[i] = input[i] * sgn * scale;
        }
    }
}