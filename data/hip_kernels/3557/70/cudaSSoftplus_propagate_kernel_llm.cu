#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cudaSSoftplus_propagate_kernel(float* x, float* y, unsigned int size)
{
    // Use shared memory for improved memory access efficiency
    __shared__ float sharedMem[1024];
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = index; i < size; i += stride) {
        float val = x[i];
        // Avoid redundant calls to exp() using shared memory
        sharedMem[threadIdx.x] = expf(val);
        __syncthreads();
        y[i] = log1pf(sharedMem[threadIdx.x]); // Use log1p for better precision
    }
}