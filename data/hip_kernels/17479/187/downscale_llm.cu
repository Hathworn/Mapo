#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    ii = ii / d1;
    x = ii;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    // Compute the global linear index
    long ii = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x;
    
    // Check if the index is within bounds
    if (ii >= no_elements) return;

    // Local variables for accumulation
    float accum = 0.0f;

    // Precompute d multiplied by scale_factor for optimization
    int s_d2 = d2 * scale_factor;
    int s_d3 = d3 * scale_factor;

    // Unroll the loop for improved performance
    for (int i = 0; i < scale_factor; ++i) {
        for (int j = 0; j < scale_factor; ++j) {
            // Calculate input index using the inverse translation function
            int ipidx = translate_idx_inv(ii, d1, s_d2, s_d3, scale_factor, i, j);
            // Accumulate results
            accum += gradOutput_data[ipidx];
        }
    }

    // Store the accumulated value to the output
    gradInput_data[ii] = accum;
}