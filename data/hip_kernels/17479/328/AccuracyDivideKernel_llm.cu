#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use atomic operation to ensure accuracy is divided correctly across threads
    atomicExch(accuracy, *accuracy / N);
}