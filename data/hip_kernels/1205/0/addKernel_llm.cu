#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global thread index for potential larger grid sizes
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure boundary check to avoid accessing out-of-bounds memory
    if (i < N) // Assuming N is defined as the total number of elements
    {
        c[i] = a[i] + b[i];
    }
}