#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* __restrict__ col_sum, int32_t* __restrict__ output, int32_t N) {
    // Calculate global index
    int32_t global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll the loop using stride
    for (int32_t i = global_idx; i < N; i += blockDim.x * gridDim.x) {
        // Use pointer offset for output
        output[blockIdx.x * N + i] = -col_sum[i];
    }
}