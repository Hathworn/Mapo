#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Use shared memory to reduce redundant reads
    __shared__ int32_t shared_row_sum;
    
    // Load row_sum[blockIdx.x] into shared memory
    if (threadIdx.x == 0) {
        shared_row_sum = row_sum[blockIdx.x];
    }
    __syncthreads();

    // Compute the offset using the loaded shared data
    for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
        output[blockIdx.x * N + i] = -shared_row_sum;
    }
}