#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Use shared memory to reduce global memory access latency
    __shared__ int64_t shared_data[256]; // Assuming blockDim.x <= 256 
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < num_elements) {
        shared_data[threadIdx.x * 3] = batch_index;
        shared_data[threadIdx.x * 3 + 1] = class_index;
        shared_data[threadIdx.x * 3 + 2] = static_cast<int64_t>(original[idx]);
        
        // Write back to global memory with coalesced access
        to_normalize[idx * 3] = shared_data[threadIdx.x * 3];
        to_normalize[idx * 3 + 1] = shared_data[threadIdx.x * 3 + 1];
        to_normalize[idx * 3 + 2] = shared_data[threadIdx.x * 3 + 2];
    }
}