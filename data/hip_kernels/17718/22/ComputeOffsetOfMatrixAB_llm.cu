#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
    int idx = blockIdx.x * N + threadIdx.x; // Calculate global index
    int stride = blockDim.x; // Define stride as blockDim
    // Loop through elements with stride
    for (int32_t i = idx; i < blockIdx.x * N + N; i += stride) {
        output[i] = K_A_B - row_sum[blockIdx.x] - col_sum[i - blockIdx.x * N];
    }
}