#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//device functions
__device__ int getGlobalIdx_1D_1D()
{
    return blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void kernel_1D_1D()
{
    int globalId = getGlobalIdx_1D_1D(); // Compute global ID only once to improve performance
    printf("Local thread ID: %i   Global thread ID: %i\n", threadIdx.x, globalId);
}