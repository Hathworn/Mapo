#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//device functions
__device__ int getGlobalIdx_1D_3D()
{
    return blockIdx.x * blockDim.x * blockDim.y * blockDim.z + 
           threadIdx.z * blockDim.y * blockDim.x + 
           threadIdx.y * blockDim.x + 
           threadIdx.x;
}

__global__ void kernel_1D_3D()
{
    // Cache block dimensions
    int blockDimX = blockDim.x;
    int blockDimY = blockDim.y;
    int blockIdxX = blockIdx.x;

    // Pre-calculate reusable offsets
    int blockOffset = blockIdxX * blockDimX * blockDimY * blockDim.z;
    int threadOffsetZ = threadIdx.z * blockDimY * blockDimX;
    int threadOffsetY = threadIdx.y * blockDimX;

    // Calculate global index from pre-calculated offsets
    int globalIdx = blockOffset + threadOffsetZ + threadOffsetY + threadIdx.x;

    printf("Local thread IDs: (%i,%i,%i)   Global thread ID: %i\n", threadIdx.x, threadIdx.y, threadIdx.z, globalIdx);
}