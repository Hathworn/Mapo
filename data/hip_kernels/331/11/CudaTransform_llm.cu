#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define DEG2RAD 0.01745329252f // Define DEG2RAD for degree to radian conversion

__device__ int getGlobalIdx_2D_2D()
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) 
                   + (threadIdx.y * blockDim.x) 
                   + threadIdx.x;
    return threadId;
}

__global__ void CudaTransform(unsigned char* dev_img, unsigned int *dev_accu, int w, int h){

    // Calculate index which this thread has to process
    unsigned int index = getGlobalIdx_2D_2D();

    // Check index is within image bounds
    if(index < (w * h)){
        // Calculate parameters
        float hough_h = (sqrtf(2.0f) * max(w, h)) / 2.0f; // Use max to decide width or height
        float center_x = w / 2.0f;
        float center_y = h / 2.0f;

        // Calculate coordinates for corresponding index in entire image
        int x = index % w;
        int y = index / w;

        if(dev_img[index] > 250){ // Check if the pixel is white (grayscale value > 250)
            for(int t = 0; t < 180; t++){ // Iterate through the parameter space from 0° to 180°
                
                // Use native trigonometric functions for performance
                float r = ( (x - center_x) * cosf(t * DEG2RAD) ) + ( (y - center_y) * sinf(t * DEG2RAD) );
                
                // Atomic increment for histogram bin
                atomicAdd(&(dev_accu[ (int)((roundf(r + hough_h) * 180.0f)) + t ]), 1);
            }
        }
    }
}