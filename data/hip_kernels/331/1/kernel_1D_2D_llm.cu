#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Macro for calculating global ID in a 1D-2D grid
#define GLOBAL_ID_1D_2D (blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x)

__global__ void kernel_1D_2D()
{
    // Print the local and global thread IDs using calculated macro
    printf("Local thread IDs: (%i,%i)   Global thread ID: %i\n", threadIdx.x, threadIdx.y, GLOBAL_ID_1D_2D);
}