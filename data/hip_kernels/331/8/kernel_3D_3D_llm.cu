#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// device functions
__device__ int getGlobalIdx_3D_3D()
{
    int blockId = blockIdx.x
                + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;

    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.z * (blockDim.x * blockDim.y))
                 + (threadIdx.y * blockDim.x)
                 + threadIdx.x;

    return threadId;
}

__global__ void kernel_3D_3D()
{
    // Load global index once for efficiency
    int globalIdx = getGlobalIdx_3D_3D();
    // Print using local function call result
    printf("Local thread IDs: (%i,%i,%i)   Global thread ID: %i\n", threadIdx.x, threadIdx.y, threadIdx.z, globalIdx);
}