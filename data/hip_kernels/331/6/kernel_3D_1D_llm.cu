#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//device functions
__device__ int getGlobalIdx_3D_1D()
{
    int blockId = blockIdx.x
                + blockIdx.y * gridDim.x
                + blockIdx.z * gridDim.x * gridDim.y;

    int threadId = blockId * blockDim.x + threadIdx.x;

    return threadId;
}

__global__ void kernel_3D_1D()
{
    // Launch fewer printf calls by using a single thread per block
    if (threadIdx.x == 0) {
        printf("Block ID: [%i, %i, %i]   First Global thread ID: %i\n", 
                blockIdx.x, blockIdx.y, blockIdx.z, getGlobalIdx_3D_1D());
    }
}