#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//device functions
__device__ int getGlobalIdx_2D_2D()
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__global__ void kernel_2D_2D()
{
    // Calculate global thread ID only once
    int globalThreadId = getGlobalIdx_2D_2D();
    
    // Use the pre-calculated global thread ID
    printf("Local thread IDs: (%i,%i)   Global thread ID: %i\n", 
           threadIdx.x, threadIdx.y, globalThreadId);
}