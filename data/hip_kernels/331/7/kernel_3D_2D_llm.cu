#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//device functions
__device__ int getGlobalIdx_3D_2D()
{
    int blockId = (blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x;
    int threadId = blockId * (blockDim.y * blockDim.x) + threadIdx.y * blockDim.x + threadIdx.x;
    return threadId;
}

__global__ void kernel_3D_2D()
{
    // Efficient global thread ID calculation with reduced arithmetic operations
    int globalThreadId = getGlobalIdx_3D_2D();
    printf("Local thread IDs: (%i,%i)   Global thread ID: %i\n", threadIdx.x, threadIdx.y, globalThreadId);
}