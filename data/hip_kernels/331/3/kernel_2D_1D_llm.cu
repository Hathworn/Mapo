#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//device functions
__device__ int getGlobalIdx_2D_1D()
{
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;
    return threadId;
}

__global__ void kernel_2D_1D()
{
    // Use variables instead of inline function calls for efficiency
    int localThreadId = threadIdx.x;
    int globalThreadId = getGlobalIdx_2D_1D();
    printf("Local thread ID: %i   Global thread ID: %i\n", localThreadId, globalThreadId);
}