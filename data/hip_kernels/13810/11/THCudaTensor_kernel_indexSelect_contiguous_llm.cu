#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexSelect_contiguous(float *tensor, float *src, long stride, float *index, long idxSize) {
    // Use constant for vectorization and warp size
    const int VT = 4;
    const int WARP_SIZE = 32;
    const int MAX_DIM_SIZE = 65535;

    // Calculate global thread index for coalesced memory access
    int idx = blockIdx.x * blockDim.y + threadIdx.y;
    int startIdx = threadIdx.x + blockIdx.y * VT * WARP_SIZE;

    while (idx < idxSize) {
        int srcIdx = ((int) index[idx] - 1) * stride;
        int targetIdx = idx * stride;

        #pragma unroll
        for (int i = 0; i < VT; i++) {
            int featureIdx = startIdx + i * WARP_SIZE;
            if (featureIdx < stride) {
                tensor[targetIdx + featureIdx] = src[srcIdx + featureIdx];
            }
        }
        idx += blockDim.y * MAX_DIM_SIZE;  // Increment index for next block
    }
}