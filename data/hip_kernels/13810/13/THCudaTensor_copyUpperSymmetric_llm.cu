#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_copyUpperSymmetric(float *input, int n, int len)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Optimize loop by avoiding unnecessary increments beyond grid size
    while (idx < len) {
        const int r = idx % n;
        const int c = idx / n;
        if (r > c) {
            input[idx] = input[r * n + c];
        }
        idx += blockDim.x * gridDim.x; // Move to next relevant index within grid
    }
}