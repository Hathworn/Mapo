#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexFill(float *tensor, long* stride, float *index, long src_nDim, int dim, long idx_size, long tensor_size, long size_dim, float val) {
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    long flat_size = tensor_size / idx_size;

    if (thread_idx < flat_size) {
        long base_index = 0;
        int leftover = thread_idx;

        // Pre-calculate the base index for the current thread
        for (int d = 0; d < src_nDim; d++) {
            if (d != dim) {
                long stride_factor = (d < dim) ? stride[d] / size_dim : stride[d];
                long coeff = leftover / stride_factor;
                leftover -= coeff * stride_factor;
                base_index += coeff * stride[d];
            }
        }

        // Update tensor values using pre-computed base index
        for (int i = 0; i < idx_size; i++) {
            tensor[base_index + (int)(index[i] - 1) * stride[dim]] = val;
        }
    }
}