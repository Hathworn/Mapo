#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const long size, const float maxnorm)
{
    __shared__ float buffer[32];
    long tx = threadIdx.x;
    long bx = blockIdx.x;
    long step = blockDim.x;
    float *row = data + size * bx;

    buffer[tx] = 0.0f;

    // Compute partial sums in parallel
    for (long i = tx; i < size; i += step) {
        buffer[tx] += powf(fabsf(row[i]), value);
    }

    // Parallel reduction to sum the buffer
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (tx < stride) {
            buffer[tx] += buffer[tx + stride];
        }
    }

    // Compute norm and renormalize if necessary
    if (tx == 0) {
        float norm = powf(buffer[0], 1.0f / value);
        if (norm > maxnorm) {
            float scale = maxnorm / (norm + 1e-7f);
            for (long i = 0; i < size; ++i) {
                row[i] *= scale;
            }
        }
    }
}