#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexSelect(float *tensor, float *src, long *src_stride, float *index, long src_nDim, int dim, long idx_size, long tensor_size, long size_dim) {
    // Calculate global thread index
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    
    // Total number of iterations required for the thread
    long flat_size = tensor_size / idx_size;

    if (thread_idx < flat_size) {
        long coeff = 0;
        int leftover;
        int targetIdx;
        int srcIdx;

        // Iterate over the index size
        for (int i = 0; i < idx_size; i++) {
            leftover = thread_idx;
            targetIdx = 0;
            srcIdx = 0;

            // Loop over the dimensions
            for (int d = 0; d < src_nDim; d++) {
                // Efficiently compute stride contributions
                if (d < dim) {
                    long stride_d = src_stride[d] / size_dim;
                    coeff = leftover / stride_d;
                    leftover %= stride_d;  // Use modulo to avoid subtraction
                    targetIdx += coeff * stride_d * idx_size;
                    srcIdx += coeff * src_stride[d];
                } else if (d > dim) {
                    coeff = leftover / src_stride[d];
                    leftover %= src_stride[d];  // Use modulo to avoid subtraction
                    targetIdx += coeff * src_stride[d];
                    srcIdx += coeff * src_stride[d];
                }
            }
            // Efficiently map the source and target index using precomputed coeffs
            tensor[targetIdx + i * src_stride[dim]] = src[srcIdx + ((int)(index[i]) - 1) * src_stride[dim]];
        }
    }
}