#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv2genericrev(float *input, float *kernel, float *output, int input_n, int input_h, int input_w, int kernel_n, int kernel_h, int kernel_w, float alpha, int stride_h, int stride_w)
{
    // output dimensions
    int output_h = input_h - (kernel_h - 1) * stride_h;
    int output_w = input_w - (kernel_w - 1) * stride_w;

    // Calculate overall thread index
    int tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;

    // one thread only sees one output
    output = output + (blockIdx.x * input_n + blockIdx.y) * output_h * output_w;

    // put the output in shared memory
    extern __shared__ float shared_output[];
    
    // Initialize shared memory
    if (tid < output_w * output_h) {
        shared_output[tid] = 0.0f;
    }
    __syncthreads();

    // convolution loop
    for (int yy = threadIdx.y; yy < output_h; yy += blockDim.y) {
        for (int xx = threadIdx.x; xx < output_w; xx += blockDim.x) {
            // Initialize sum accumulation
            float sum = 0.0f;
            float *input_p = input + (blockIdx.y * input_n + threadIdx.z) * input_h * input_w + yy * stride_h * input_w + xx * stride_w;
            float *kernel_p = kernel + (blockIdx.x * kernel_n + threadIdx.z) * kernel_w * kernel_h;
            for (int ky = 0; ky < kernel_h; ky++) {
                #pragma unroll
                for (int kx = 0; kx < kernel_w; kx++) {
                    sum += input_p[kx] * kernel_p[kx];
                }
                input_p += input_w;
                kernel_p += kernel_w;
            }
            shared_output[yy * output_w + xx] += sum;
        }
    }

    // Wait for all threads to finish writing to shared memory
    __syncthreads();

    // Batch reduction and writing back to global memory
    if (threadIdx.y == 0 && threadIdx.x == 0) {
        for (int i = tid; i < output_w * output_h; i += blockDim.x * blockDim.y * blockDim.z) {
            output[i] += alpha * shared_output[i];
        }
    }
}