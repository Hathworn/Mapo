#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution(uint8_t *inData, uint8_t *outData, int width, int height, float *kernel, int kwidth, int kheight, float ksum)
{
    // Calculate global x and y coordinates for thread
    int gx = blockIdx.x * blockDim.x + threadIdx.x;
    int gy = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure the thread is within bounds
    if (gx < width && gy < height) {
        int rx = (kwidth - 1) / 2;
        int ry = (kheight - 1) / 2;

        float sum = 0.0f;
        
        // Traverse the kernel area
        for (int y = -ry; y <= ry; ++y) {
            int cy = max(0, min(height - 1, gy + y));
            for (int x = -rx; x <= rx; ++x) {
                int cx = max(0, min(width - 1, gx + x));
                sum += inData[cx + cy * width] * kernel[(x + rx) + (y + ry) * kwidth];
            }
        }

        // Clamp sum value to the valid range [0, 255] and divide by kernel sum
        outData[gx + gy * width] = (uint8_t) max(0.0f, min(255.0f, sum / ksum));
    }
}