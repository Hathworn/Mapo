#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void erosion2(uint8_t *inData, uint8_t *outData, int radiusX, int radiusY, int width, int height)
{
    __shared__ uint8_t localData[TILE_SIZE * TILE_SIZE];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int gx = blockIdx.x * blockDim.x + tx;
    int gy = blockIdx.y * blockDim.y + ty;

    // Cache input to shared memory with boundary checks
    int sharedIndex = TILE_SIZE * (radiusY + ty) + radiusX + tx;
    
    int x = min(max(gx, 0), width - 1);
    int y = min(max(gy, 0), height - 1);

    localData[sharedIndex] = inData[width * y + x];
    __syncthreads();

    // Initialize search boundaries
    int x1 = max(tx - radiusX, 0);
    int x2 = min(tx + radiusX, blockDim.x - 1);
    int y1 = max(ty - radiusY, 0);
    int y2 = min(ty + radiusY, blockDim.y - 1);

    uint8_t minimum = 255;
    // Traverse the neighborhood
    for (int j = y1; j <= y2; ++j) {
        for (int i = x1; i <= x2; ++i) {
            minimum = min(minimum, localData[TILE_SIZE * (radiusY + j) + radiusX + i]);
        }
    }

    // Write the minimum value to the output
    if (gx < width && gy < height) {
        outData[width * gy + gx] = minimum;
    }
}