#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void erosion(uint8_t *inData, uint8_t *outData, int radiusX, int radiusY, int width, int height)
{
    int gx = blockIdx.x * blockDim.x + threadIdx.x;
    int gy = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (gx >= width || gy >= height) { // Early exit for out-of-bound threads
        return;
    }

    int x1 = max(0, gx - radiusX);        // Clamp x1 to 0
    int x2 = min(width - 1, gx + radiusX); // Clamp x2 to width - 1
    int y1 = max(0, gy - radiusY);        // Clamp y1 to 0
    int y2 = min(height - 1, gy + radiusY); // Clamp y2 to height - 1

    uint8_t minimum = 255;

    for (int y = y1; y <= y2; ++y) {
        for (int x = x1; x <= x2; ++x) {
            minimum = min(minimum, inData[width * y + x]);
        }
    }

    outData[width * gy + gx] = minimum;
}