#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//32x32
#define NTHREADS_X 16
#define NTHREADS_Y 32
#define THREADS_PER_BLOCK NTHREADS_X * NTHREADS_Y

/* 
 * Error checking macro for HIP function calls 
 * Based on: http://stackoverflow.com/a/14038590
 */
__global__ void matrix_mul(int *a, int *b, int *c, int a_ncolumns, int c_nlines, int c_ncolumns)
{
    // Calculate global column and row indices
    int column = blockIdx.x * blockDim.x + threadIdx.x;
    int line = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within bounds of result matrix
    if (column >= c_ncolumns || line >= c_nlines)
        return;

    // Keep intermediate sum for dot product
    int sum = 0;
    int beginA = a_ncolumns * line;
    int beginB = column;

    // Perform dot product calculation for line and column
    for (int i = 0; i < a_ncolumns; i++)
    {
        sum += a[beginA + i] * b[i * c_ncolumns + beginB];
    }

    // Write the result back to the matrix C
    c[line * c_ncolumns + column] = sum;
}