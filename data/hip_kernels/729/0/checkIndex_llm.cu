#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void checkIndex(void) {
    // Optimize by storing frequently accessed values in local variables
    int tx = threadIdx.x, ty = threadIdx.y, tz = threadIdx.z;
    int bx = blockIdx.x, by = blockIdx.y, bz = blockIdx.z;
    int bdx = blockDim.x, bdy = blockDim.y, bdz = blockDim.z;
    int gdx = gridDim.x, gdy = gridDim.y, gdz = gridDim.z;

    printf("threadIdx: (%d,%d,%d) blockIdx: (%d, %d, %d) blockDim: (%d, %d, %d) gridDim: (%d, %d, %d)\n",
           tx, ty, tz, bx, by, bz, bdx, bdy, bdz, gdx, gdy, gdz);
}