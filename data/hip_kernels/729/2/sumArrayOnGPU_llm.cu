#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArrayOnGPU(float *A, float *B, float *C) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread ID
    C[i] = A[i] + B[i]; // Perform vector addition
}