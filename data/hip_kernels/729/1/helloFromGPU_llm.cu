#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU(){
    // Use a variable to avoid multiple calls to threadIdx.x
    int idx = threadIdx.x;
    if(idx == 5){
        printf("Hello World from GPU! thread %d\n", idx);
    }
}