#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate the global index once
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Efficient bound check and processing in loop
    for (; i < n; i += blockDim.x * gridDim.x) {
        c[i] += a[i] * b[i];
    }
}