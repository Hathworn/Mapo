#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float dc_val = dc[i];
        float s_val = s[i];
        if (da) da[i] += dc_val * s_val; // Update da if non-null
        db[i] += dc_val * (1 - s_val);  // Update db with pre-computed value
        ds[i] += dc_val * (a[i] - b[i]); // Combine operations on ds for efficiency
    }
}