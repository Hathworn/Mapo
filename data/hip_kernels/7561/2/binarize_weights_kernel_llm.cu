#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

extern "C" {
double get_time_point();
void start_timer();
void stop_timer();
double get_time();
void stop_timer_and_show();
void stop_timer_and_show_name(char *name);
void show_total_time();
}

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    // Calculate unique global thread index
    int f = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index does not exceed number of weight sets
    if (f < n) {
        float mean = 0.0f;

        // Use a single loop to calculate mean directly
        for (int i = 0; i < size; ++i) {
            float w = weights[f * size + i];
            mean += fabsf(w);
            binary[f * size + i] = (w > 0) ? 1.0f : -1.0f;
        }
        mean /= size;

        // Multiply binary results with mean in a single pass
        for (int i = 0; i < size; ++i) {
            binary[f * size + i] *= mean;
        }
    }
}