#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Use shared memory for better memory access coalescing
    __shared__ float partialSum[256]; // Adjust size according to blockDim.x

    partialSum[threadIdx.x] = 0.0f;

    // Unroll loop for efficient memory access
    for (int k = threadIdx.x; k < n; k += blockDim.x) {
        partialSum[threadIdx.x] += x[k * groups + i];
    }
    __syncthreads();

    // Reduce within block
    if (threadIdx.x == 0) {
        for (int j = 1; j < blockDim.x; j++) {
            partialSum[0] += partialSum[j];
        }
        sum[i] = partialSum[0];
    }
}