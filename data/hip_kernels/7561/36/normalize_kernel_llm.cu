#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x; 
    if (index >= N) return; 

    // Calculate filter index
    int f = (index / spatial) % filters;

    // Perform normalization
    float var_sqrt = sqrtf(variance[f]) + 1e-6f; // Reduced precision constant
    x[index] = (x[index] - mean[f]) / var_sqrt;
}