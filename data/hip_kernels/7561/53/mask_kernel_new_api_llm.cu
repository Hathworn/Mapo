#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mask_kernel_new_api(int n, float *x, float mask_num, float *mask, float val)
{
    // Use blockIdx.x directly with gridDim.x for simplified indexing
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Correct loop iteration for thread-compact execution
    for (int idx = i; idx < n; idx += blockDim.x * gridDim.x) {
        if (mask[idx] == mask_num) {
            x[idx] = val;
        }
    }
}