#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate the global thread ID
    if (id >= n) return;

    int j = id % in_w;
    int i = (id / in_w) % in_h;
    int k = (id / in_w / in_h) % in_c;
    int b = id / in_w / in_h / in_c;

    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;
    int area = (size - 1) / stride;

    float d = 0;
    for (int l = -area; l <= area; ++l) {  // Combine loop conditions
        for (int m = -area; m <= area; ++m) {  // Combine loop conditions
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            // Simplify index calculations into a single conditional expression
            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h) {
                int out_index = out_w + w * (out_h + h * (k + c * b));
                if (indexes[out_index] == id) {
                    d += delta[out_index];
                }
            }
        }
    }
    prev_delta[id] += d;
}