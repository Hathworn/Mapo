#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

extern "C" {
double get_time_point();
void start_timer();
void stop_timer();
double get_time();
void stop_timer_and_show();
void stop_timer_and_show_name(char *name);
void show_total_time();
}

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (i < n) // Changed condition to ensure valid range
    {
        binary[i] = (x[i] >= 0) ? 1.0f : -1.0f; // Explicit float literals
    }
}