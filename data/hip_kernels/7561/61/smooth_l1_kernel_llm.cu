#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize thread indexing in 1D
    if(i < n){
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff); // Use fast single-precision absolute function
        if(abs_val < 1.0f) { // Use consistent float literals
            error[i] = diff * diff;
            delta[i] = diff;
        }
        else {
            error[i] = 2.0f * abs_val - 1.0f; // Use consistent float literals
            delta[i] = copysignf(1.0f, diff); // Use copysignf for cleaner conditional
        }
    }
}