#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Calculate scale factor
    float scale = 1.0f / (batch * spatial);
    
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Return if thread ID exceeds the number of filters
    if (i >= filters) return;
    
    // Initialize mean for the current filter
    float sum = 0.0f;
    
    // Use shared memory to reduce global memory access latency
    extern __shared__ float sharedData[];
    
    // Loop through batches and spatial elements
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += x[index];
        }
    }
    
    // Store the result in the global mean array
    mean[i] = sum * scale;
}