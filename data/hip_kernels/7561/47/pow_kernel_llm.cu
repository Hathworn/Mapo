#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    int gridSize = blockDim.x * gridDim.x;           // Total number of threads across the grid
    while (idx < N) {                                // Loop through all necessary elements
        Y[idx * INCY] = powf(X[idx * INCX], ALPHA);  // Perform the power operation
        idx += gridSize;                             // Move to the next element this thread should handle
    }
}