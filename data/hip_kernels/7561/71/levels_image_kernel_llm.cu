#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x;
    float s = hsv.y;
    float v = hsv.z;

    float r, g, b;
    int index = (int)floorf(h) % 6; // Normalize h to [0, 6)
    float f = h - index;
    float p = v * (1 - s);
    float q = v * (1 - s * f);
    float t = v * (1 - s * (1 - f));

    // Assign r, g, b based on the index
    r = (index == 0) ? v : (index == 1) ? q : (index == 2) ? p : (index == 3) ? p : (index == 4) ? t : v;
    g = (index == 0) ? t : (index == 1) ? v : (index == 2) ? v : (index == 3) ? q : (index == 4) ? p : p;
    b = (index == 0) ? p : (index == 1) ? p : (index == 2) ? t : (index == 3) ? v : (index == 4) ? v : q;

    return make_float3(fminf(fmaxf(r, 0.0f), 1.0f), fminf(fmaxf(g, 0.0f), 1.0f), fminf(fmaxf(b, 0.0f), 1.0f));
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x;
    float g = rgb.y;
    float b = rgb.z;

    float max = fmaxf(r, fmaxf(g, b));
    float min = fminf(r, fminf(g, b));
    float delta = max - min;
    float h = 0, s = (max != 0) ? (delta / max) : 0, v = max;

    if (delta != 0) {
        if (r == max) h = (g - b) / delta + (g < b ? 6.0f : 0.0f);
        else if (g == max) h = (b - r) / delta + 2.0f;
        else if (b == max) h = (r - g) / delta + 4.0f;
        h /= 6.0f; // Normalize h to [0, 1)
    } else {
        h = 0; // h is undefined when s is 0
    }

    return make_float3(h, s, v);
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return;

    int x = id % w;
    id /= w;
    int y = id % h;
    id /= h;

    float rshift = rand[0];
    float gshift = rand[1];
    float bshift = rand[2];
    float r0 = rand[8 * id + 0];
    float r1 = rand[8 * id + 1];
    float r2 = rand[8 * id + 2];
    float r3 = rand[8 * id + 3];

    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > 0.5f) ? 1.0f / saturation : saturation;
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > 0.5f) ? 1.0f / exposure : exposure;

    size_t offset = id * h * w * 3;
    image += offset;
    float r = image[x + w * (y + h * 0)];
    float g = image[x + w * (y + h * 1)];
    float b = image[x + w * (y + h * 2)];
    float3 rgb = make_float3(r, g, b);

    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    }

    image[x + w * (y + h * 0)] = rgb.x * scale + translate + (rshift - 0.5f) * shift;
    image[x + w * (y + h * 1)] = rgb.y * scale + translate + (gshift - 0.5f) * shift;
    image[x + w * (y + h * 2)] = rgb.z * scale + translate + (bshift - 0.5f) * shift;
}