#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using shared memory and thread-level parallelism
__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        float t = truth[i];
        float p = max(pred[i], 1e-8f); // Avoid log(0) with clamp
        error[i] = t ? -__logf(p) : 0.0f; // Use fast math intrinsic
        delta[i] = t - p;
    }
}