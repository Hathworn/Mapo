#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

extern "C" {
double get_time_point();
void start_timer();
void stop_timer();
double get_time();
void stop_timer_and_show();
void stop_timer_and_show_name(char *name);
void show_total_time();
}

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    // Calculate the global index for each thread
    int s = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

    // Return if global index exceeds size
    if (s >= size) return;

    // Optimizing mean calculation using reduction
    float mean = 0;
    for(int i = 0; i < n; ++i) {
        mean += fabs(input[i * size + s]);
    }
    mean = mean / n;

    // Assign binary values based on mean
    for(int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}