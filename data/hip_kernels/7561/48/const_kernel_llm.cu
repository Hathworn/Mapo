#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Compute the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Avoid calculating unnecessary values and improve warp efficiency
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x)
    {
        X[idx * INCX] = ALPHA;
    }
}