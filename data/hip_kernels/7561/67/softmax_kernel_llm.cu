#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    float sum = 0;
    float largest = -INFINITY;
    // Use shared memory to cache input values for better performance
    extern __shared__ float shared_input[];
    
    int tid = threadIdx.x;
    if (tid < n) {
        shared_input[tid] = input[tid];
        __syncthreads();
        
        // Calculate the largest in parallel
        atomicMax(&largest, shared_input[tid]);
        __syncthreads();
        
        float e = exp(shared_input[tid] / temp - largest / temp);
        
        // Use atomic operations to calculate the sum in parallel
        atomicAdd(&sum, e);
        __syncthreads();
        
        output[tid] = e / sum;
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if(b >= batch) return;
    
    // Launch with dynamic shared memory allocation
    softmax_device<<<1, n, n * sizeof(float)>>>(n, input + b * offset, temp, output + b * offset);
}