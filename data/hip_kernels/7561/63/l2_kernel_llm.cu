#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified 1D grid stride
    int gridSize = blockDim.x * gridDim.x; // Calculate the total stride based on grid size
    while (i < n) { // Use a loop to iterate over large data
        float diff = truth[i] - pred[i];
        error[i] = diff * diff;
        delta[i] = diff;
        i += gridSize; // Move to the next batch of data within the grid
    }
}