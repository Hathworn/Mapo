#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;
    int i1, i2;

    // Using shared memory to reduce global memory access
    extern __shared__ float shared_output[];

    // Load data into shared memory
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            i1 = b * size * n + f1 * size + i;
            i2 = b * size * n + f2 * size + i;
            shared_output[threadIdx.x * 2] = output[i1];
            shared_output[threadIdx.x * 2 + 1] = output[i2];
            __syncthreads();

            float val1 = shared_output[threadIdx.x * 2];
            float val2 = shared_output[threadIdx.x * 2 + 1];
            sum += val1 * val2;
            norm1 += val1 * val1;
            norm2 += val2 * val2;
            __syncthreads();
        }
    }

    // Avoid repeated division by norm1 and norm2
    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float inv_norm = 1.0f / (norm1 * norm2);
    sum *= inv_norm;

    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            i1 = b * size * n + f1 * size + i;
            i2 = b * size * n + f2 * size + i;
            float shared_val1 = output[i1];
            float shared_val2 = output[i2];
            shared_output[threadIdx.x * 2] = shared_val1;
            shared_output[threadIdx.x * 2 + 1] = shared_val2;
            __syncthreads();

            delta[i1] += -scale * sum * shared_output[threadIdx.x * 2 + 1] * inv_norm;
            delta[i2] += -scale * sum * shared_output[threadIdx.x * 2] * inv_norm;
            __syncthreads();
        }
    }
}