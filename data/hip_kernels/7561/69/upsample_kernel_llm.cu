#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Optimized index computation
    int out_w = (i % (w * stride));
    int ih = (i / (w * stride));
    int out_h = (ih % (h * stride));
    int ic = (ih / (h * stride)) % c;
    int b = (ih / (h * stride)) / c;

    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_index = b * w * h * c + ic * w * h + in_h * w + in_w;

    // Conditional handling
    if (forward) {
        out[i] += scale * x[in_index];
    } else {
        atomicAdd(&x[in_index], scale * out[i]);
    }
}