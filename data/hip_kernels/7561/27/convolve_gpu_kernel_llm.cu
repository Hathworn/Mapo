#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolve_gpu_kernel(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n, int size, int pad)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int x = index % in_w;
    int index2 = index / in_w;
    int y = index2 % in_h;
    int fil = index2 / in_h;

    // Early exit if fil index is out of range
    if (fil >= n) return;

    // Calculate output index and initialize sum
    int output_index = fil * in_w * in_h + y * in_w + x;
    float sum = 0.0f;

    // Loop over channels
    for (int chan = 0; chan < in_c; ++chan)
    {
        int input_pre_index = chan * in_w * in_h;
        int weights_pre_index = fil * in_c * size * size + chan * size * size;

        // Loop over filter height and width
        for (int f_y = 0; f_y < size; ++f_y)
        {
            int input_y = y + f_y - pad;
            if (input_y < 0 || input_y >= in_h) continue;  // Skip invalid y
            
            for (int f_x = 0; f_x < size; ++f_x)
            {
                int input_x = x + f_x - pad;
                if (input_x < 0 || input_x >= in_w) continue;  // Skip invalid x

                int input_index = input_pre_index + input_y * in_w + input_x;
                int weights_index = weights_pre_index + f_y * size + f_x;

                // Sum the product of input and weights
                sum += input[input_index] * weights[weights_index];
            }
        }
    }

    // Update the output at the current index
    output[output_index] = sum;
}