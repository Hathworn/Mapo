#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Avoid out-of-bound memory access
    if (i >= N) return;
    
    // Calculate indices
    int in_s = i % spatial;
    int b = i / (spatial * layers);
    int in_c = (i / spatial) % layers;

    // Optimize memory accesses
    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;

    // Conditional write to output
    out[forward ? i2 : i1] = forward ? x[i1] : x[i2];
}