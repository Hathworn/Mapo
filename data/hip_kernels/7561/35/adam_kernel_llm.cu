#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;
    
    // Precompute reused variables outside of the per-thread scope
    float sqrt_1_minus_B2_t = sqrtf(1.F - powf(B2, t));
    float inv_1_minus_B1_t = 1.F / (1.F - powf(B1, t));

    // Update position with precomputed variables
    x[index] = x[index] - (rate * sqrt_1_minus_B2_t * inv_1_minus_B1_t * m[index] / (sqrtf(v[index]) + eps));
}