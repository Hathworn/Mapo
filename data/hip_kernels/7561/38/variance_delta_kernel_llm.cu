#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float var_epsilon_inv = -.5f * powf(variance[i] + .000001f, -1.5f);  // Precompute common expression
    float vari_delta = 0;  // Use local variable to reduce global memory access

    for(int j = 0; j < batch; ++j){
        for(int k = 0; k < spatial; ++k){
            int index = j * filters * spatial + i * spatial + k;
            vari_delta += delta[index] * (x[index] - mean[i]);  // Accumulate variance delta contributions
        }
    }

    variance_delta[i] = vari_delta * var_epsilon_inv;  // Apply precomputed factor
}
```
