#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int total_elements = w * h;
    int k = id % c;
    int b = id / c;
    int out_index = k + c * b;
    int in_offset = h * w * (k + b * c);

    float sum = 0.0f;
    for(int i = 0; i < total_elements; ++i){
        sum += input[in_offset + i];
    }
    output[out_index] = sum / total_elements;
}