#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access pattern and reduce grid strides
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = gridDim.x * blockDim.x; // Calculate grid stride
    for (; i < N; i += gridStride) { // Use strided loop for memory coalescing
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}