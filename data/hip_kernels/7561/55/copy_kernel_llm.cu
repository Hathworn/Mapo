#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N,  float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate unique thread index using 1D grid for better memory access pattern
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure threads within valid range
    if (i < N) {
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}