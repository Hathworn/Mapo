#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + 
            blockIdx.y * gridDim.x * blockDim.x;

    // Ensure within bounds and apply mask conditionally
    if (i < n) {
        float mask_val = mask[i]; // Avoid redundant global memory access
        if (mask_val == mask_num) {
            x[i] = mask_num;
        }
    }
}