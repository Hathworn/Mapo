#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void activate_array_leaky_kernel(float *x, int n)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within the array bounds
    if (index < n) {
        // Utilize inline conditional operator for potential speedup
        x[index] = (x[index] > 0) ? x[index] : x[index] * 0.1f;
    }
}