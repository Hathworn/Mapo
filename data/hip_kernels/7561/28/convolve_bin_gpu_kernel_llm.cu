#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __host__ static inline uint8_t xnor_bit1(uint8_t a, uint8_t b) {
    return ~(a ^ b) & 0b1;
}

__device__ __host__ static inline unsigned char get_bit(unsigned char const* const src, size_t index) {
    size_t src_i = index / 8;
    int src_shift = index % 8;
    unsigned char val = (src[src_i] & (1 << src_shift)) > 0;
    return val;
}

__global__ void convolve_bin_gpu_kernel(float *input, float *weights, float *output, int in_w, int in_h, int in_c, int n, int size, int pad, int new_lda, float *mean_arr_gpu) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int x = index % in_w;
    int index2 = index / in_w;
    int y = index2 % in_h;
    int fil = index2 / in_h;

    if (fil < n) {
        int const output_index = fil * in_w * in_h + y * in_w + x;
        int sum = 0;
        int good_val = 0;
        
        for (int chan = 0; chan < in_c; ++chan) {
            int const weights_pre_index = fil * new_lda + chan * size * size;
            int const input_pre_index = chan * in_w * in_h;
            
            // Use shared memory for weights and inputs
            __shared__ uint8_t shared_weights[BLOCK_SIZE];
            __shared__ uint8_t shared_input[BLOCK_SIZE];

            // Efficient data reading into shared memory
            for (int f_y = 0; f_y < size; ++f_y) {
                int input_y = y + f_y - pad;
                for (int f_x = 0; f_x < size; ++f_x) {
                    int input_x = x + f_x - pad;
                    if (input_y < 0 || input_x < 0 || input_y >= in_h || input_x >= in_w) continue;

                    int input_index = input_pre_index + input_y * in_w + input_x;
                    int weights_index = weights_pre_index + f_y * size + f_x;

                    shared_input[threadIdx.x] = get_bit((uint8_t *)input, input_index);
                    shared_weights[threadIdx.x] = get_bit((uint8_t *)weights, weights_index);
                    __syncthreads();  // Synchronize to ensure all threads have read the necessary data

                    uint8_t in_bit = shared_input[threadIdx.x];
                    uint8_t w_bit = shared_weights[threadIdx.x];
                    int res = xnor_bit1(in_bit, w_bit);
                    sum += res;
                    good_val++;
                }
            }
        }
        sum = sum - (good_val - sum);
        output[output_index] = sum * mean_arr_gpu[fil];
    }
}