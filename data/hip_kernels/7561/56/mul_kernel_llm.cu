#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify thread index calculation for 1D grid
    if(i < N) 
    {
        Y[i * INCY] *= X[i * INCX]; // Perform operation on each element
    }
}