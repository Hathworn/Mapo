#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    // Calculate global thread ID more clearly
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a stride loop to handle larger data sizes efficiently
    for (; id < size; id += blockDim.x * gridDim.x) {
        input[id] = (rand[id] < prob) ? 0 : input[id] * scale;
    }
}