#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Shared memory to reduce global memory bandwidth
    extern __shared__ float shared_delta[];

    // Initialize shared memory for accumulation
    float local_mean_delta = 0.0f;

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            local_mean_delta += delta[index];
        }
    }

    // Write the result back to global memory with normalization
    mean_delta[i] = local_mean_delta * (-1.F / sqrtf(variance[i] + 0.000001f));
}