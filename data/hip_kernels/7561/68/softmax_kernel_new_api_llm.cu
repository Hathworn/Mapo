#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device_new_api(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Calculate the largest element for numerical stability
    for (i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = (val > largest) ? val : largest;
    }

    // Calculate the exponentials and their sum
    for (i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp);
        sum += e;
        output[i * stride] = e;
    }

    // Normalize by dividing by the sum of exponentials
    for (i = 0; i < n; ++i) {
        output[i * stride] /= sum;
    }
}

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Calculate the largest element for numerical stability
    for (i = 0; i < n; ++i) {
        float val = input[i];
        largest = (val > largest) ? val : largest;
    }

    // Calculate the exponentials and their sum
    for (i = 0; i < n; ++i) {
        float e = expf((input[i] - largest) / temp);
        sum += e;
        output[i] = e;
    }

    // Normalize by dividing by the sum of exponentials
    for (i = 0; i < n; ++i) {
        output[i] /= sum;
    }
}

__global__ void softmax_kernel_new_api(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (id >= batch * groups) return;

    int b = id / groups;
    int g = id % groups;

    // Compute softmax for each group in the batch
    softmax_device_new_api(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}