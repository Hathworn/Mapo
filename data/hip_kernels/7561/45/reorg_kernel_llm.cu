#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (idx >= N) return;
    
    int b = (idx / (w * h * c)) % batch;
    int in_c = (idx / (w * h)) % c;
    int in_h = (idx / w) % h;
    int in_w = idx % w;

    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    
    int out_index = b * (out_c * h * stride * w * stride) + c2 * (h * stride * w * stride) + h2 * w * stride + w2;

    if (forward)
        out[out_index] = x[idx];
    else
        out[idx] = x[out_index];
}