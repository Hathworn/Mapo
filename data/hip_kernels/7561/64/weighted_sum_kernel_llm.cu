#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Check index within bounds
    if(i < n) {
        // Compute weighted sum
        float b_value = (b != nullptr) ? b[i] : 0.0f;
        c[i] = s[i] * a[i] + (1.0f - s[i]) * b_value;
    }
}