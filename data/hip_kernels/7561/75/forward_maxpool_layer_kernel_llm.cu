#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int c = in_c;
    
    // Use blockIdx.z for additional dimension indices to increase parallelism
    int id = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    id = id * blockDim.x + threadIdx.x;

    if (id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    int out_index = j + w * (i + h * (k + c * b));
    float max = -INFINITY;
    int max_i = -1;
    
    // Unroll loops to reduce loop control overhead
    #pragma unroll
    for (int l = 0; l < size; ++l) {
        #pragma unroll
        for (int m = 0; m < size; ++m) {
            int cur_h = h_offset + i * stride + l;
            int cur_w = w_offset + j * stride + m;
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            int valid = (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w);
            float val = valid ? input[index] : -INFINITY;

            // Use fmaxf() for potential slight performance gain in selecting the maximum value
            if(val > max) {
                max = val;
                max_i = index;
            }
        }
    }
    
    output[out_index] = max;
    indexes[out_index] = max_i;
}