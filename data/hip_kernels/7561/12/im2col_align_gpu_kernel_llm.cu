#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void im2col_align_gpu_kernel(const int n, const float* data_im, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_col, const int bit_align)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;

    for (; index < n; index += total_threads) {  // Reduced register pressure by using total_threads
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;

        float* data_col_ptr = &data_col[channel_out * bit_align + h_out * width_col + w_out];
        const float* data_im_ptr = data_im + (channel_in * height + h_in) * width + w_in;

        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                float val = (h >= 0 && w >= 0 && h < height && w < width) ? data_im_ptr[i * width + j] : 0;

                int pre_out_index = index % (width_col * height_col);
                int out_index = (channel_out + i * ksize + j) * bit_align + pre_out_index;
                data_col[out_index] = val;

                data_col_ptr += bit_align;  // Improved data memory offset calculation
            }
        }
    }
}