#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

extern "C" {
double get_time_point();
void start_timer();
void stop_timer();
double get_time();
void stop_timer_and_show();
void stop_timer_and_show_name(char *name);
void show_total_time();
}

__global__ void set_zero_kernel(float *src, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a loop to efficiently set zero when more threads than elements
    for (; i < size; i += blockDim.x * gridDim.x) {
        src[i] = 0;
    }
}