#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1.F / (batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float var = 0.0f; // Use a register for intermediate variance

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            float diff = x[index] - mean[i];
            var += diff * diff; // Use multiplication instead of powf
        }
    }

    variance[i] = var * scale; // Assign calculated variance back to global memory
}