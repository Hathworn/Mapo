#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x; // Optimize index calculation
    if (index >= N) return;

    int spatialInv = 1 / (spatial * batch); // Precompute frequently used value
    int f = (index/spatial)%filters;

    float varInv = 1.F / (sqrtf(variance[f]) + .000001f); // Precompute inverse variance
    float delta_value = delta[index];
    float value_diff = 2.0f * variance_delta[f] * (x[index] - mean[f]);

    delta[index] = delta_value * varInv + value_diff * spatialInv + mean_delta[f] * spatialInv;
}