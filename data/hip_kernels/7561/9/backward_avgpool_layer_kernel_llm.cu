#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (int idx = id; idx < n; idx += stride) { // Loop unrolling for better occupancy
        int k = idx % c;
        int b = idx / c;
        
        int out_index = k + c * b;
        float shared_out_delta = out_delta[out_index] / (w * h); // Store division result in shared variable
        
        for (int i = 0; i < w * h; ++i) {
            int in_index = i + h * w * (k + b * c);
            atomicAdd(&in_delta[in_index], shared_out_delta); // Use atomicAdd for concurrent writes
        }
    }
}