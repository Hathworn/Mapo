#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use int to calculate the unique id of the thread in 1D 
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure that only valid threads process calculations
    if(global_id < N) {
        // Use shared memory for storing ALPHA*ALPHA to avoid repetitive calculations
        __shared__ float alpha_squared;
        if (threadIdx.x == 0) {
            alpha_squared = ALPHA * ALPHA;
        }
        __syncthreads();

        float element_squared = X[global_id * INCX] * X[global_id * INCX];
        if(element_squared < alpha_squared) {
            X[global_id * INCX] = 0;
        }
    }
}