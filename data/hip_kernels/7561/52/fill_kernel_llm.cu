#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index

    // Use grid-stride loop for better utilization
    for (; i < N; i += blockDim.x * gridDim.x)
    {
        X[i * INCX] = ALPHA;  // Set the value at the correct index
    }
}