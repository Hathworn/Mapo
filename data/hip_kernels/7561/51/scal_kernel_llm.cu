#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop with a stride to handle more elements per thread
    for (; i < N; i += gridDim.x * blockDim.x) {
        X[i * INCX] *= ALPHA;
    }
}