#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure coalesced access by using INCX
    int index = i * INCX; 

    // Perform operation if within bounds
    if(index < N) {
        float x_val = X[index];
        
        // Apply constraints using ALPHA
        X[index] = fminf(ALPHA, fmaxf(-ALPHA, x_val));
    }
}