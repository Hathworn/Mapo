#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    if(offset < size) {
        // Using shared memory for improved access speed
        __shared__ float shared_bias;
        if (threadIdx.x == 0) {
            shared_bias = biases[filter];
        }
        __syncthreads();
        output[(batch*n+filter)*size + offset] *= shared_bias;
    }
}