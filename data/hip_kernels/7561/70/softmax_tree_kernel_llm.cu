#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device_new_api(float *input, int n, float temp, int stride, float *output) {
    float sum = 0;
    float largest = -INFINITY;
    // Use parallel reduction to find the largest element
    for (int i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = max(val, largest);
    }
    __syncthreads(); // Ensure all threads have computed the largest

    // Compute exponential values and sum them up
    for (int i = 0; i < n; ++i) {
        float e = expf(input[i * stride] / temp - largest / temp);
        sum += e;
        output[i * stride] = e;
    }
    __syncthreads(); // Ensure sum is computed correctly

    // Normalize the results to get probability distribution
    for (int i = 0; i < n; ++i) {
        output[i * stride] /= sum;
    }
}

__device__ void softmax_device(int n, float *input, float temp, float *output) {
    float sum = 0;
    float largest = -INFINITY;
    // Use parallel reduction to find the largest element
    for (int i = 0; i < n; ++i) {
        float val = input[i];
        largest = max(val, largest);
    }
    __syncthreads();

    // Compute exponential values and sum them up
    for (int i = 0; i < n; ++i) {
        float e = exp(input[i] / temp - largest / temp);
        sum += e;
        output[i] = e;
    }
    __syncthreads();

    // Normalize the results to get the probability distribution
    for (int i = 0; i < n; ++i) {
        output[i] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset) {
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;
    int s = id % spatial;
    id = id / spatial;
    int g = id % groups;
    int b = id / groups;
    int goff = group_offset[g] * spatial;
    int boff = b * stride;
    softmax_device_new_api(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}