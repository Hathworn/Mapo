#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__shared__ int smem[324];

__global__ void convolution1Kernel(int *dst, int *src, int rows, int cols, int *filter) {
    // Calculate global position
    int posx = threadIdx.x + blockIdx.x * blockDim.x;
    int posy = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Check boundaries
    if (posx > 0 && posy > 0 && posx < rows - 1 && posy < cols - 1) {
        
        // Initialize the destination value
        int dstVal = 0;

        // Loop over filter
        for (int k = 0; k < 3; ++k) {
            for (int l = 0; l < 3; ++l) {
                dstVal += src[(posy + k - 1) * cols + (posx + l - 1)] * filter[k * 3 + l];
            }
        }
        
        // Write back to global memory
        dst[posy * cols + posx] = dstVal;
    }
}