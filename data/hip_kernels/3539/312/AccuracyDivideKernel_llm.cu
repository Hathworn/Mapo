#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
  // Calculate the thread index
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  // Use one thread to perform the division
  if (index == 0) {
    *accuracy /= N;
  }
}