#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void frame_encode_staircase(int *data, int *pcm, int param_k, int param_m, int w_f, int packet_size)
{
    int ps = packet_size / sizeof(int);

    // Use blockIdx.x to process different parts of data concurrently
    int block_offset = blockIdx.x * ps;
    int x = threadIdx.x;

    for (int index = param_k; index < param_k + param_m - 1; index++)
    {
        int offset = x;

        while (offset < ps)
        {
            // Optimize memory access patterns
            int dataIndex1 = (index + 1) * ps + block_offset + offset;
            int dataIndex2 = index * ps + block_offset + offset;
            data[dataIndex1] ^= data[dataIndex2];
            offset += blockDim.x;
        }
    }
}
```
