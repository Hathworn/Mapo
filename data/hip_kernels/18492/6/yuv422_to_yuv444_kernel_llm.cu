#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void yuv422_to_yuv444_kernel(const void * src, void * out, int pix_count) {
    // Calculate global thread index
    const int global_idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Each thread processes two pixels (4 YUV422-packed bytes)
    const int out_idx = global_idx * 3; // Output index offset for 3 YUV444 pixels
    const int in_idx = global_idx * 2;  // Input index offset for 2 YUV422 pixels

    // Skip if out of bounds
    if (in_idx >= pix_count) {
        return;
    }

    uchar4 *src_data = (uchar4 *)src;
    uchar4 *out_data = (uchar4 *)out;

    // Fetch two sets of YUV422 pixels
    uchar4 pix12 = src_data[in_idx];
    uchar4 pix34 = src_data[in_idx + 1];

    // Output 3 YUV444 pixels from the 2 YUV422 pixels
    out_data[out_idx]     = make_uchar4(pix12.y, pix12.x, pix12.z, pix12.w);   // First Y, U, V, A
    out_data[out_idx + 1] = make_uchar4(pix12.x, pix12.z, pix34.y, pix34.x);   // Second Y, V, Y, U
    out_data[out_idx + 2] = make_uchar4(pix34.z, pix34.w, pix34.x, pix34.z);   // Third V, A, Y, V
}