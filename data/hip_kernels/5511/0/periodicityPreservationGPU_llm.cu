#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BUFSIZE 64
#define BLOCK_SIZE 16

// Periodicity Preservation retains our periodicity
__global__ void periodicityPreservationGPU(int N, char *cells)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use threads for rows adaptation
    if (i >= 1 && i <= N) {
        // Copy first real row to bottom extra row
        cells[(N+2)*(N+1)+i] = cells[(N+2)+i];
        
        // Copy last real row to top extra row
        cells[i] = cells[(N+2)*N + i];
    }

    // Ensure all threads complete row operations before column operations
    __syncthreads();

    // Use threads for columns adaptation
    if (i <= N+1) {
        // Copy first real column to right last extra column
        cells[i*(N+2)+N+1] = cells[i*(N+2)+1];
        
        // Copy last real column to left last extra column
        cells[i*(N+2)] = cells[i*(N+2) + N];
    }
}