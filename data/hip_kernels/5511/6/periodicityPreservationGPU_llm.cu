#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void periodicityPreservationGPU(int N, char *cells)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Optimize rows with parallel threads
    if (idx >= 1 && idx <= N) {
        // Copy first real row to bottom extra row
        cells[(N+2)*(N+1)+idx] = cells[(N+2)+idx];
        // Copy last real row to top extra row
        cells[idx] = cells[(N+2)*N + idx];
    }
    
    // Optimize columns with parallel threads
    if (idx >= 0 && idx <= N+1) {
        // Copy first real column to right last extra column
        cells[idx*(N+2)+N+1] = cells[idx*(N+2)+1];
        // Copy last real column to left last extra column
        cells[idx*(N+2)] = cells[idx*(N+2) + N];
    }
}