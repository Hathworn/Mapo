#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void evovle_kernel_shared(int N, char *oldGen, char *newGen, int *allzeros, int *change)
{
    // Global
    int ix = (blockDim.x - 2) * blockIdx.x + threadIdx.x;
    int iy = (blockDim.y - 2) * blockIdx.y + threadIdx.y;
    int id = ix * (N+2) + iy;

    int i = threadIdx.x;
    int j = threadIdx.y;
    int neighbors;

    // Declare the shared memory on a per block level
    __shared__ char oldGen_shared[BLOCK_SIZE][BLOCK_SIZE];

    // Copy cells into shared memory
    if (ix <= N+1 && iy <= N+1)
        oldGen_shared[i][j] = oldGen[id];

    // Sync threads on block
    __syncthreads();

    if (ix <= N && iy <= N) {
        // Avoid boundary conflicts
        if(i != 0 && i != (blockDim.y-1) && j != 0 && j != (blockDim.x-1)) {
            // Calculate neighbors more efficiently using shared memory
            neighbors = oldGen_shared[i+1][j] + oldGen_shared[i-1][j]
                      + oldGen_shared[i][j+1] + oldGen_shared[i][j-1]
                      + oldGen_shared[i+1][j+1] + oldGen_shared[i-1][j-1]
                      + oldGen_shared[i-1][j+1] + oldGen_shared[i+1][j-1];

            char cell = oldGen_shared[i][j];
            newGen[id] = (neighbors == 3 || (neighbors == 2 && cell));

            // Atomic operations to avoid race conditions
            if (newGen[id] != 0) atomicAdd(allzeros, 1);
            if (newGen[id] != oldGen[id]) atomicAdd(change, 1);
        }
    }
}