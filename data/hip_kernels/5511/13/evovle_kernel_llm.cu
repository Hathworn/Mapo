#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void evovle_kernel(int N, char *oldGen, char *newGen, int *allzeros, int *change)
{
    // Efficient indexing for the thread
    int ix = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int iy = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int id = ix * (N+2) + iy;

    if (ix <= N && iy <= N) {
        int neighbors = oldGen[id+(N+2)] + oldGen[id-(N+2)]    // lower upper 
                      + oldGen[id+1] + oldGen[id-1]           // right left 
                      + oldGen[id+(N+3)] + oldGen[id-(N+3)]   // diagonals 
                      + oldGen[id-(N+1)] + oldGen[id+(N+1)];

        char cell = oldGen[id];
        newGen[id] = (neighbors == 3 || (neighbors == 2 && cell)); // Compute new cell state

        // Atomic operations for safe increment
        if (newGen[id] != 0) atomicAdd(allzeros, 1);          // Counter for live cells
        if (newGen[id] != oldGen[id]) atomicAdd(change, 1);   // Counter for changes
    }
}