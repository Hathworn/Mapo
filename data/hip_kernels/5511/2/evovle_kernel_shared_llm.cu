#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BUFSIZE 64
#define BLOCK_SIZE 16

__global__ void evovle_kernel_shared(int N, char *oldGen, char *newGen, int *allzeros, int *change) {
    // Calculate global indices
    int ix = (blockDim.x - 2) * blockIdx.x + threadIdx.x;
    int iy = (blockDim.y - 2) * blockIdx.y + threadIdx.y;
    int id = ix * (N+2) + iy;

    // Local indices for shared memory
    int i = threadIdx.x;
    int j = threadIdx.y;

    // Shared memory declaration for current block
    __shared__ char oldGen_shared[BLOCK_SIZE][BLOCK_SIZE];

    // Copy cells into shared memory
    if (ix <= N+1 && iy <= N+1)
        oldGen_shared[i][j] = oldGen[id];

    // Synchronize threads to ensure shared memory is populated
    __syncthreads();

    // Main computation: Check boundaries to avoid accessing out-of-bounds memory
    if (ix <= N && iy <= N && i > 0 && i < blockDim.y - 1 && j > 0 && j < blockDim.x - 1) {
        // Calculate neighbors
        int neighbors = oldGen_shared[i+1][j] + oldGen_shared[i-1][j] +
                        oldGen_shared[i][j+1] + oldGen_shared[i][j-1] +
                        oldGen_shared[i+1][j+1] + oldGen_shared[i-1][j-1] +
                        oldGen_shared[i-1][j+1] + oldGen_shared[i+1][j-1];

        // Update cell
        char cell = oldGen_shared[i][j];
        newGen[id] = (neighbors == 3 || (neighbors == 2 && cell));

        // Check for changes
        if (newGen[id] != 0) atomicAdd(allzeros, 1);
        if (newGen[id] != oldGen[id]) atomicAdd(change, 1);
    }
}