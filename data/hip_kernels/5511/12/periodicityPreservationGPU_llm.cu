#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void periodicityPreservationGPU(int N, char *cells)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize rows operation using index
    if (idx > 0 && idx <= N) {
        // Copy first real row to bottom extra row
        cells[(N+2)*(N+1)+idx] = cells[(N+2)+idx];
        
        // Copy last real row to top extra row
        cells[idx] = cells[(N+2)*N + idx];
    }

    // Optimize cols operation using index
    if (idx <= N+1) {
        // Copy first real column to right last extra column
        cells[idx*(N+2)+N+1] = cells[idx*(N+2)+1];
        
        // Copy last real column to left last extra column
        cells[idx*(N+2)] = cells[idx*(N+2) + N];
    }
}