#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BUFSIZE 64
#define BLOCK_SIZE 16

// Optimized kernel for evolve function
__global__ void evovle_kernel(int N, char *oldGen, char *newGen, int *allzeros, int *change)
{
    // Achieve indexing on 2D blocks
    int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
    // Thread calculates its global id
    int id = ix * (N + 2) + iy;

    if (ix <= N && iy <= N) {
        // Reduce redundant calculations by using temporary variables
        int rowOffset = (N + 2);
        int neighbors = oldGen[id + rowOffset] + oldGen[id - rowOffset] // lower upper
                      + oldGen[id + 1] + oldGen[id - 1]               // right left
                      + oldGen[id + rowOffset + 1] + oldGen[id - rowOffset - 1] // diagonals
                      + oldGen[id - rowOffset + 1] + oldGen[id + rowOffset - 1];

        char cell = oldGen[id];
        newGen[id] = neighbors == 3 || (neighbors == 2 && cell); // Fill in the cells

        // Terminating Checkings
        if (newGen[id] != 0) atomicAdd(allzeros, 1); // Check if all cells are dead
        if (newGen[id] != oldGen[id]) atomicAdd(change, 1); // Check if life stayed the same
    }
}