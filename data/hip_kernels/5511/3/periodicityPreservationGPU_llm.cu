#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void periodicityPreservationGPU(int N, char *cells)
{
    // Calculate once and reuse
    int stride = N + 2;
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Optimize using parallel threads
    if (tid <= N)
    {
        // Copy first real row to bottom extra row
        cells[stride * (N + 1) + tid + 1] = cells[stride + tid + 1];

        // Copy last real row to top extra row
        cells[tid + 1] = cells[stride * N + tid + 1];
    }
    
    if (tid <= N + 1)
    {
        // Copy first real column to right last extra column
        cells[tid * stride + N + 1] = cells[tid * stride + 1];

        // Copy last real column to left last extra column
        cells[tid * stride] = cells[tid * stride + N];
    }
}