#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void evovle_kernel(int N, char *oldGen, char *newGen, int *allzeros, int *change)
{
    // Achieve indexing on 2D blocks
    int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
    // Thread calculates its global id
    int id = ix * (N+2) + iy;

    if (ix <= N && iy <= N) {
        int neighbors = oldGen[id+(N+2)] + oldGen[id-(N+2)] // lower upper
                      + oldGen[id+1] + oldGen[id-1]         // right left
                      + oldGen[id+(N+3)] + oldGen[id-(N+3)] // diagonals
                      + oldGen[id-(N+1)] + oldGen[id+(N+1)];
        
        // Conditional assignment to reduce branches
        char cell = oldGen[id];
        char newCell = (neighbors == 3 || (neighbors == 2 && cell));

        newGen[id] = newCell; // Fill in the cells

        // Atomic operations for concurrent update when the condition is met
        if (newCell != 0) atomicAdd(allzeros, 1); 
        if (newCell != cell) atomicAdd(change, 1);
    }
}