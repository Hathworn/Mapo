#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void evolve_kernel(int N, char *oldGen, char *newGen, int *allzeros, int *change) {
    // Compute unique global index for each thread
    int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
    int id = ix * (N+2) + iy;

    // Initialize neighbors count
    int neighbors = 0;

    if (ix <= N && iy <= N) {
        // Calculate number of alive neighbors
        neighbors += oldGen[id + (N+2)] + oldGen[id - (N+2)]   // lower and upper 
                  + oldGen[id + 1] + oldGen[id - 1]             // right and left
                  + oldGen[id + (N+3)] + oldGen[id - (N+3)]     // diagonals
                  + oldGen[id - (N+1)] + oldGen[id + (N+1)];

        // Determine new cell state based on neighbors and current state
        newGen[id] = neighbors == 3 || (neighbors == 2 && oldGen[id]);

        // Atomic increment to avoid race conditions
        atomicAdd(allzeros, newGen[id] != 0 ? 1 : 0);        // Check if all cells are dead
        atomicAdd(change, newGen[id] != oldGen[id] ? 1 : 0); // Check if life changed
    }
}