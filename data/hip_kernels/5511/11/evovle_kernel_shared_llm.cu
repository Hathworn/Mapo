#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void evovle_kernel_shared(int N, char *oldGen, char *newGen, int *allzeros, int *change)
{
    // Global indices with adjusted block boundaries
    int ix = (blockDim.x - 2) * blockIdx.x + threadIdx.x;
    int iy = (blockDim.y - 2) * blockIdx.y + threadIdx.y;
    int id = ix * (N+2) + iy;

    int i = threadIdx.x;
    int j = threadIdx.y;
    int neighbors;

    // Shared memory for storing sub-grid
    __shared__ char oldGen_shared[BLOCK_SIZE][BLOCK_SIZE];

    // Load cells into shared memory 
    if (ix <= N+1 && iy <= N+1)
        oldGen_shared[i][j] = oldGen[id];

    // Synchronize to make sure all cells are loaded into shared memory
    __syncthreads();

    // Check if within bounds of grid excluding ghost cells
    if (ix <= N && iy <= N) {
        // Ensure only active threads within boundary procceed
        if(i != 0 && i != (blockDim.x-1) && j != 0 && j != (blockDim.y-1)) {

            // Compute the number of neighbors from shared memory
            neighbors = oldGen_shared[i+1][j] + oldGen_shared[i-1][j]     // vertical
                      + oldGen_shared[i][j+1] + oldGen_shared[i][j-1]     // horizontal
                      + oldGen_shared[i+1][j+1] + oldGen_shared[i-1][j-1] // diagonals
                      + oldGen_shared[i-1][j+1] + oldGen_shared[i+1][j-1];

            char cell = oldGen_shared[i][j];
            newGen[id] = neighbors == 3 || (neighbors == 2 && cell); // Update condition

            // Update change/watching conditions atomically
            if (newGen[id] != 0) atomicAdd(allzeros, 1);
            if (newGen[id] != oldGen[id]) atomicAdd(change, 1);
        }
    }
}