#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void evovle_kernel(int N, char *oldGen, char *newGen, int *allzeros, int *change) {
    // 2D Indexing
    int ix = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int iy = blockDim.y * blockIdx.y + threadIdx.y + 1;
    // Calculate global id
    int id = ix * (N+2) + iy;

    if (ix <= N && iy <= N) {
        int neighbors;
        // Shared memory for neighboring calculations optimization
        __shared__ char sharedOldGen[32][32]; // Assuming blockDim.x and blockDim.y <= 30 for simplicity
        
        // Load data into shared memory, include halo for neighbor access
        sharedOldGen[threadIdx.x + 1][threadIdx.y + 1] = oldGen[id];
        if (threadIdx.x == 0)
            sharedOldGen[0][threadIdx.y + 1] = oldGen[id - (N+2)];
        if (threadIdx.x == blockDim.x - 1)
            sharedOldGen[blockDim.x + 1][threadIdx.y + 1] = oldGen[id + (N+2)];
        if (threadIdx.y == 0)
            sharedOldGen[threadIdx.x + 1][0] = oldGen[id - 1];
        if (threadIdx.y == blockDim.y - 1)
            sharedOldGen[threadIdx.x + 1][blockDim.y + 1] = oldGen[id + 1];

        __syncthreads();

        // Calculate number of neighbors
        neighbors = sharedOldGen[threadIdx.x + 2][threadIdx.y + 1] + sharedOldGen[threadIdx.x][threadIdx.y + 1]  // lower upper
                  + sharedOldGen[threadIdx.x + 1][threadIdx.y + 2] + sharedOldGen[threadIdx.x + 1][threadIdx.y]  // right left
                  + sharedOldGen[threadIdx.x + 2][threadIdx.y + 2] + sharedOldGen[threadIdx.x][threadIdx.y]      // diagonals
                  + sharedOldGen[threadIdx.x][threadIdx.y + 2] + sharedOldGen[threadIdx.x + 2][threadIdx.y]; 

        // Cell update
        char cell = sharedOldGen[threadIdx.x + 1][threadIdx.y + 1];
        newGen[id] = neighbors == 3 || (neighbors == 2 && cell);

        // Terminating check
        if (newGen[id] != 0) atomicAdd(allzeros, 1);  // Use atomic operation for global memory
        if (newGen[id] != oldGen[id]) atomicAdd(change, 1); // Use atomic operation for global memory
    }
}