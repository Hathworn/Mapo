#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void periodicityPreservationGPU(int N, char* cells) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Efficient check to prevent excess threads
    if (i >= N + 2) return;

    // Copy first real row to bottom extra row
    if (i >= 1 && i <= N) {
        cells[(N + 2) * (N + 1) + i] = cells[(N + 2) + i];
        cells[i] = cells[(N + 2) * N + i];
    }

    // Copy first real column to right last extra column
    cells[i * (N + 2) + N + 1] = cells[i * (N + 2) + 1];
    cells[i * (N + 2)] = cells[i * (N + 2) + N];
}