#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void evovle_kernel_shared(int N, char *oldGen, char *newGen, int *allzeros, int *change) {
    // Global indices for accessing the array
    int ix = (blockDim.x - 2) * blockIdx.x + threadIdx.x;
    int iy = (blockDim.y - 2) * blockIdx.y + threadIdx.y;
    int id = ix * (N+2) + iy;

    // Local thread indices for accessing shared memory
    int i = threadIdx.x;
    int j = threadIdx.y;
    int neighbors;

    // Declare shared memory
    __shared__ char oldGen_shared[BLOCK_SIZE][BLOCK_SIZE];

    // Copy cells to shared memory with boundary check
    if (ix <= N+1 && iy <= N+1) 
        oldGen_shared[i][j] = oldGen[id];

    // Synchronize threads within a block
    __syncthreads();

    // Check the block and thread boundaries
    if (ix <= N && iy <= N && i > 0 && i < blockDim.y-1 && j > 0 && j < blockDim.x-1) {
        // Calculate the number of neighbors
        neighbors = oldGen_shared[i+1][j] + oldGen_shared[i-1][j]
                  + oldGen_shared[i][j+1] + oldGen_shared[i][j-1]
                  + oldGen_shared[i+1][j+1] + oldGen_shared[i-1][j-1]
                  + oldGen_shared[i-1][j+1] + oldGen_shared[i+1][j-1];

        char cell = oldGen_shared[i][j];
        newGen[id] = (neighbors == 3 || (neighbors == 2 && cell));

        // Check any changes or all zero condition for termination
        atomicAdd(allzeros, newGen[id] != 0); 
        atomicAdd(change, newGen[id] != oldGen[id]);
    }
}