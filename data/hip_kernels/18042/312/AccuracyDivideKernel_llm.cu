#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load the accuracy value into a register to reduce global memory access
    float acc = *accuracy;
    
    // Perform the division using a register
    acc /= N;
    
    // Write the result back to global memory
    *accuracy = acc;
}