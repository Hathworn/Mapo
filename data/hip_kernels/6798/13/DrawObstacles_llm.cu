#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DrawObstacles(uchar4 *ptr, int* indices, int size) {
    // Calculate global thread index
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;

    // Loop through elements using stride pattern for better memory access
    for (int i = thread_id; i < size; i += blockDim.x * gridDim.x) {
        int index = indices[i];
        ptr[index] = make_uchar4(0, 0, 0, 255); // Use efficient built-in function
    }
}