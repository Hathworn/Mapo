#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory for better performance if N is large
    __shared__ float shared_accuracy;
    
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy;
    }
    __syncthreads();
    
    if (threadIdx.x == 0) {
        // Perform the division on one thread for global memory updating
        shared_accuracy /= N;
        *accuracy = shared_accuracy;
    }
}