#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void propose_kernel(int64_t *out, int64_t *proposal, const int64_t *rowptr, const int64_t *col, int64_t numel) {
    const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_idx < numel && out[thread_idx] == -1) { // Combine condition checks
        bool has_unmatched_neighbor = false;

        for (int64_t i = rowptr[thread_idx]; i < rowptr[thread_idx + 1]; i++) {
            int64_t v = col[i]; // Declare v as int64_t

            if (out[v] < 0) {
                has_unmatched_neighbor = true; // Unmatched neighbor found.
            }

            if (out[v] == -2) {
                proposal[thread_idx] = v; // Propose to first red neighbor.
                break;
            }
        }

        if (!has_unmatched_neighbor) {
            out[thread_idx] = thread_idx; // Assign thread_idx directly to out
        }
    }
}