#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void respond_kernel(int64_t *out, const int64_t *proposal, const int64_t *rowptr, const int64_t *col, int64_t numel) {
    const int64_t thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_idx < numel) {
        if (out[thread_idx] != -2) return;  // Only visit red nodes.

        bool has_unmatched_neighbor = false;
        int64_t start_idx = rowptr[thread_idx];
        int64_t end_idx = rowptr[thread_idx + 1];

        // Unrolling for efficiency, assuming row degree is small
        for (int64_t i = start_idx; i < end_idx; i++) {
            int64_t v = col[i];

            if (out[v] < 0) has_unmatched_neighbor = true; // Unmatched neighbor found.

            if (out[v] == -1 && proposal[v] == thread_idx) {
                // Match first blue neighbor v which proposed to u.
                int64_t min_thread_v = min(thread_idx, v);
                out[thread_idx] = min_thread_v;
                out[v] = min_thread_v;
                break;
            }
        }

        if (!has_unmatched_neighbor) {
            out[thread_idx] = thread_idx;
        }
    }
}