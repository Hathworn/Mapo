#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void calcpi(int threads, long n, double *results) {
    int rank = threadIdx.x + blockIdx.x * blockDim.x; // Use global index for rank
    if (rank >= threads) return; // Boundary check to avoid out-of-bounds access

    results[rank] = 0.0;
    double step = 1.0 / n;
    double x = 0.0;

    long lower = rank * n / threads;
    long upper = (rank + 1) * n / threads;

    for (long i = lower; i < upper; i++) {
        x = (i + 0.5) * step;
        results[rank] += 4.0 / (1.0 + x * x);
    }
    results[rank] *= step; // Apply step scaling to result
}