#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* /usr/local/cuda/bin/nvcc -gencode arch=compute_20,code=compute_20 -o fw_kernel.ptx -ptx fw_kernel.cu
*/

extern "C" {


}
__global__ void fw(float *adj_array, int *next_array, int k, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    if (i < N && j < N)
    {
        float check = adj_array[j * N + k] + adj_array[k * N + i];
        if (check < adj_array[j * N + i])
        {
            // Update only if a shorter path is found
            adj_array[j * N + i] = check;
            next_array[j * N + i] = next_array[j * N + k];
        }
    }
}