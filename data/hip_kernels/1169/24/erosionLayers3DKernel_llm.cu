#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void erosionLayers3DKernel( unsigned short *d_dst, unsigned short *d_src, int w, int h, int d, int kernel_radius )
{
    __shared__ unsigned short smem[ER_LAYERS_BLOCKDIM_X][ER_LAYERS_BLOCKDIM_Y][(ER_LAYERS_RESULT_STEPS + 2 * ER_LAYERS_HALO_STEPS) * ER_LAYERS_BLOCKDIM_Z + 1];
    unsigned short *smem_thread = smem[threadIdx.x][threadIdx.y];

    // Calculate base positions for source/destination pointers
    const int baseX = blockIdx.x * ER_LAYERS_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ER_LAYERS_BLOCKDIM_Y + threadIdx.y;
    const int baseZ = (blockIdx.z * ER_LAYERS_RESULT_STEPS - ER_LAYERS_HALO_STEPS) * ER_LAYERS_BLOCKDIM_Z + threadIdx.z;
    d_src += (baseZ * h + baseY) * w + baseX;
    d_dst += (baseZ * h + baseY) * w + baseX;

    const int pitch = w * h;

    // Load main data into shared memory
    #pragma unroll
    for (int i = ER_LAYERS_HALO_STEPS; i < ER_LAYERS_HALO_STEPS + ER_LAYERS_RESULT_STEPS; i++) {
        smem_thread[threadIdx.z + i * ER_LAYERS_BLOCKDIM_Z] = d_src[i * ER_LAYERS_BLOCKDIM_Z * pitch];
    }

    // Load upper halo into shared memory
    #pragma unroll
    for (int i = 0; i < ER_LAYERS_HALO_STEPS; i++) {
        smem_thread[threadIdx.z + i * ER_LAYERS_BLOCKDIM_Z] = (baseZ + i * ER_LAYERS_BLOCKDIM_Z >= 0) ? d_src[i * ER_LAYERS_BLOCKDIM_Z * pitch] : USHRT_MAX;
    }

    // Load lower halo into shared memory
    #pragma unroll
    for (int i = ER_LAYERS_HALO_STEPS + ER_LAYERS_RESULT_STEPS; i < ER_LAYERS_HALO_STEPS + ER_LAYERS_RESULT_STEPS + ER_LAYERS_HALO_STEPS; i++) {
        smem_thread[threadIdx.z + i * ER_LAYERS_BLOCKDIM_Z]= (baseZ + i * ER_LAYERS_BLOCKDIM_Z < d) ? d_src[i * ER_LAYERS_BLOCKDIM_Z * pitch] : USHRT_MAX;
    }

    // Synchronize to make sure data is ready for processing
    __syncthreads();

    // Compute erosion results and store them
    #pragma unroll
    for (int i = ER_LAYERS_HALO_STEPS; i < ER_LAYERS_HALO_STEPS + ER_LAYERS_RESULT_STEPS; i++) {
        unsigned short *smem_kern = &smem_thread[threadIdx.z + i * ER_LAYERS_BLOCKDIM_Z - kernel_radius];
        unsigned short val = USHRT_MAX; // Initialize with maximum value

        #pragma unroll
        for (int j = 0; j <= 2 * kernel_radius; j++) {
            val = min(val, smem_kern[j]);
        }

        d_dst[i * ER_LAYERS_BLOCKDIM_Z * pitch] = val;
    }
}