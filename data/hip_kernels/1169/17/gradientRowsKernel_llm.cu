#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gradientRowsKernel(float *d_Dst, float *d_Src, int imageW, int imageH, int imageD) 
{
    __shared__ float s_Data[ROWS_GRAD_BLOCKDIM_Z][ROWS_GRAD_BLOCKDIM_Y][(ROWS_GRAD_RESULT_STEPS + 2 * ROWS_GRAD_HALO_STEPS) * ROWS_GRAD_BLOCKDIM_X];

    // Offset to the left halo edge
    const int baseX = (blockIdx.x * ROWS_GRAD_RESULT_STEPS - ROWS_GRAD_HALO_STEPS) * ROWS_GRAD_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * ROWS_GRAD_BLOCKDIM_Y + threadIdx.y;
    const int baseZ = blockIdx.z * ROWS_GRAD_BLOCKDIM_Z + threadIdx.z;

    d_Src += (baseZ * imageH + baseY) * imageW + baseX;
    d_Dst += (baseZ * imageH + baseY) * imageW + baseX;

    // Load data including halos
    #pragma unroll
    for (int i = 0; i < ROWS_GRAD_RESULT_STEPS + 2 * ROWS_GRAD_HALO_STEPS; i++) {
        int offsetX = baseX + i * ROWS_GRAD_BLOCKDIM_X;
        s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * ROWS_GRAD_BLOCKDIM_X] = (offsetX >= 0 && offsetX < imageW) ? d_Src[i * ROWS_GRAD_BLOCKDIM_X] : 0.0f;
    }

    __syncthreads();

    // Compute and store results
    #pragma unroll
    for (int i = ROWS_GRAD_HALO_STEPS; i < ROWS_GRAD_HALO_STEPS + ROWS_GRAD_RESULT_STEPS; i++) {
        float sum = s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * ROWS_GRAD_BLOCKDIM_X + 1]
                  - s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * ROWS_GRAD_BLOCKDIM_X - 1];
        sum *= 0.5f;
        d_Dst[i * ROWS_GRAD_BLOCKDIM_X] = sum;
    }
}