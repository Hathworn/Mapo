#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void DivideKernel ( float *d_dst, unsigned short *d_denom ) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x; // Use both block and thread indices
    if (idx < gridDim.x * blockDim.x) { // Boundary check to avoid out of bounds
        d_dst[idx] /= d_denom[idx]; // Perform division
    }
}