#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
* CCL3D.cu
*/

#define CCL_BLOCK_SIZE_X 8
#define CCL_BLOCK_SIZE_Y 8
#define CCL_BLOCK_SIZE_Z 8

__device__ int d_isNotDone;

__global__ void scanLabels(int* labels, int w, int h, int d) {
    // Calculate the index only if within bounds
    const int x = blockIdx.x * CCL_BLOCK_SIZE_X + threadIdx.x;
    if (x >= w) return;
    const int y = blockIdx.y * CCL_BLOCK_SIZE_Y + threadIdx.y;
    if (y >= h) return;
    const int z = blockIdx.z * CCL_BLOCK_SIZE_Z + threadIdx.z;
    if (z >= d) return;
    const int index = (z*h + y)*w + x;

    const int Z1 = w * h;
    const int Y1 = w;

    int lcur = labels[index];
    if (lcur) {
        int lmin = index; // Initialize lmin to the current index to hold the possible minimum label

        // Use local variables to reduce repeated memory access
        int lne, pos;
        for (int Zdif = -Z1; Zdif <= Z1; Zdif += Z1) {
            for (int Ydif = -Y1; Ydif <= Y1; Ydif += Y1) {
                for (int Xdif = -1; Xdif <= 1; Xdif += 1) {
                    pos = index + Zdif + Ydif + Xdif;
                    if (pos >= 0 && pos < w*h*d) {
                        lne = labels[pos];
                        if (lne && lne < lmin) {
                            lmin = lne;
                        }
                    }
                }
            }
        }

        if (lmin < lcur) {
            // Ensure atomic update to avoid race condition
            atomicMin(&labels[lcur], lmin);
            d_isNotDone = 1;
        }
    }
}