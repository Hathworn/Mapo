#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeSubtract3DKernel(float *img_src, const float *img_sub, const int width, const int height, const int depth, float normalizer) {
    const int baseX = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x for flexibility
    const int baseY = blockIdx.y * blockDim.y + threadIdx.y; // Use blockDim.y for flexibility
    const int baseZ = blockIdx.z * blockDim.z + threadIdx.z; // Use blockDim.z for flexibility

    if (baseX < width && baseY < height && baseZ < depth) { // Ensure indices are within bounds
        const int idx = (baseZ * height + baseY) * width + baseX;
        img_src[idx] = (img_src[idx] - img_sub[idx]) * normalizer;
    }
}