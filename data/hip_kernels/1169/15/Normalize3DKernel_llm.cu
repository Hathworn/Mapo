#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Normalize3DKernel (const unsigned short *d_src, const float *d_erosion, const float *d_dilation, float *d_dst, float min_intensity, const int width, const int height, const int depth) {
    // Calculate the 3D index
    const int baseX = blockIdx.x * blockDim.x + threadIdx.x;
    const int baseY = blockIdx.y * blockDim.y + threadIdx.y;
    const int baseZ = blockIdx.z * blockDim.z + threadIdx.z;

    // bounds check
    if (baseX >= width || baseY >= height || baseZ >= depth) return;

    // Compute linear index
    const int idx = (baseZ * height + baseY) * width + baseX;
    const float intensity = static_cast<float>(d_src[idx]);

    // Normalize if intensity is above threshold
    d_dst[idx] = (intensity >= min_intensity) ? (intensity - d_erosion[idx]) / (d_dilation[idx] - d_erosion[idx]) : 0.0f;
}