#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Copy3DKernel (const unsigned short *d_src, float *d_dst, float min_intensity, const int width, const int height, const int depth) {
    // Calculate global index for current thread
    const int baseX = blockIdx.x * blockDim.x + threadIdx.x;
    const int baseY = blockIdx.y * blockDim.y + threadIdx.y;
    const int baseZ = blockIdx.z * blockDim.z + threadIdx.z;

    // Return if thread is out of bounds
    if (baseX >= width || baseY >= height || baseZ >= depth) return;

    // Calculate 1D index in the 3D array
    const int idx = (baseZ * height + baseY) * width + baseX;

    // Perform intensity check and value copy
    const float intensity = (float)d_src[idx];
    d_dst[idx] = (intensity >= min_intensity) ? intensity : 0;
}