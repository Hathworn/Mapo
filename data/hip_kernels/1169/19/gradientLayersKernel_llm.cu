#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gradientLayersKernel(float *d_Dst, float *d_Src, int imageW, int imageH, int imageD) {
    // Use shared memory to cache data
    __shared__ float s_Data[LAYERS_GRAD_BLOCKDIM_X][LAYERS_GRAD_BLOCKDIM_Y][(LAYERS_GRAD_RESULT_STEPS + 2 * LAYERS_GRAD_HALO_STEPS) * LAYERS_GRAD_BLOCKDIM_Z + 1];

    // Offset to the upper halo edge
    const int baseX = blockIdx.x * LAYERS_GRAD_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * LAYERS_GRAD_BLOCKDIM_Y + threadIdx.y;
    const int baseZ = (blockIdx.z * LAYERS_GRAD_RESULT_STEPS - LAYERS_GRAD_HALO_STEPS) * LAYERS_GRAD_BLOCKDIM_Z + threadIdx.z;
    d_Src += (baseZ * imageH + baseY) * imageW + baseX;
    d_Dst += (baseZ * imageH + baseY) * imageW + baseX;

    const int pitch = imageW * imageH;

    // Load data into shared memory
    #pragma unroll
    for (int i = 0; i < LAYERS_GRAD_RESULT_STEPS + 2 * LAYERS_GRAD_HALO_STEPS; i++) {
        int zCoord = baseZ + i * LAYERS_GRAD_BLOCKDIM_Z;
        s_Data[threadIdx.x][threadIdx.y][threadIdx.z + i * LAYERS_GRAD_BLOCKDIM_Z] = 
            (zCoord >= 0 && zCoord < imageD) ? d_Src[i * LAYERS_GRAD_BLOCKDIM_Z * pitch] : 0.0f;
    }

    // Compute and store results
    __syncthreads();
    #pragma unroll
    for (int i = LAYERS_GRAD_HALO_STEPS; i < LAYERS_GRAD_HALO_STEPS + LAYERS_GRAD_RESULT_STEPS; i++) {
        float sum = s_Data[threadIdx.x][threadIdx.y][threadIdx.z + i * LAYERS_GRAD_BLOCKDIM_Z + 1]
                  - s_Data[threadIdx.x][threadIdx.y][threadIdx.z + i * LAYERS_GRAD_BLOCKDIM_Z - 1];
        sum *= 0.5f;

        d_Dst[i * LAYERS_GRAD_BLOCKDIM_Z * pitch] = sum;
    }
}