#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void HessianPositiveDefiniteKernel( char *d_hessian_pd, float *d_Src, int imageW, int imageH, int imageD )
{
    __shared__ float s_Data[HES_BLOCKDIM_Z+2][HES_BLOCKDIM_Y+2][(HES_RESULT_STEPS + 2 * HES_HALO_STEPS) * HES_BLOCKDIM_X];

    // Calculate thread's base coordinates in the grid
    const int baseX = (blockIdx.x * HES_RESULT_STEPS - HES_HALO_STEPS) * HES_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * HES_BLOCKDIM_Y + threadIdx.y - 1;
    const int baseZ = blockIdx.z * HES_BLOCKDIM_Z + threadIdx.z - 1;
    const int idx = (baseZ * imageH + baseY) * imageW + baseX;

    d_Src += idx;  
    d_hessian_pd += idx;

    // Check if thread is within the valid working section
    if(baseZ < 0 || baseZ >= imageD || baseY < 0 || baseY >= imageH) {
        for (int i = 0; i < HES_HALO_STEPS + HES_RESULT_STEPS + HES_HALO_STEPS; i++) {
            s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * HES_BLOCKDIM_X] = 0;
        }
        return;
    }

    // Load main data into shared memory
    #pragma unroll
    for (int i = HES_HALO_STEPS; i < HES_HALO_STEPS + HES_RESULT_STEPS; i++) {
        s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * HES_BLOCKDIM_X] = d_Src[i * HES_BLOCKDIM_X];
    }

    // Load left halo data
    #pragma unroll
    for (int i = 0; i < HES_HALO_STEPS; i++) {
        s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * HES_BLOCKDIM_X] = (baseX + i * HES_BLOCKDIM_X >= 0) ? d_Src[i * HES_BLOCKDIM_X] : 0.0f;
    }

    // Load right halo data
    #pragma unroll
    for (int i = HES_HALO_STEPS + HES_RESULT_STEPS; i < HES_HALO_STEPS + HES_RESULT_STEPS + HES_HALO_STEPS; i++) {
        s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * HES_BLOCKDIM_X] = (baseX + i * HES_BLOCKDIM_X < imageW) ? d_Src[i * HES_BLOCKDIM_X] : 0.0f;
    }

    // yz edge is no need to compute
    if (threadIdx.z == 0 || threadIdx.z == HES_BLOCKDIM_Z + 1 || threadIdx.y == 0 || threadIdx.y == HES_BLOCKDIM_Y + 1)
        return;

    __syncthreads();

    // Compute and store results
    #pragma unroll
    for (int i = HES_HALO_STEPS; i < HES_HALO_STEPS + HES_RESULT_STEPS; i++)
    {
        float xx, xy, xz, yy, yz, zz;
        xx = s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * HES_BLOCKDIM_X - 1]
           + s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * HES_BLOCKDIM_X + 1]
           - s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * HES_BLOCKDIM_X] * 2;
        xy = s_Data[threadIdx.z][threadIdx.y + 1][threadIdx.x + i * HES_BLOCKDIM_X + 1]
           + s_Data[threadIdx.z][threadIdx.y - 1][threadIdx.x + i * HES_BLOCKDIM_X - 1]
           - s_Data[threadIdx.z][threadIdx.y + 1][threadIdx.x + i * HES_BLOCKDIM_X - 1]
           - s_Data[threadIdx.z][threadIdx.y - 1][threadIdx.x + i * HES_BLOCKDIM_X + 1];
        xz = s_Data[threadIdx.z + 1][threadIdx.y][threadIdx.x + i * HES_BLOCKDIM_X + 1]
           + s_Data[threadIdx.z - 1][threadIdx.y][threadIdx.x + i * HES_BLOCKDIM_X - 1]
           - s_Data[threadIdx.z + 1][threadIdx.y][threadIdx.x + i * HES_BLOCKDIM_X - 1]
           - s_Data[threadIdx.z - 1][threadIdx.y][threadIdx.x + i * HES_BLOCKDIM_X + 1];
        yy = s_Data[threadIdx.z][threadIdx.y + 1][threadIdx.x + i * HES_BLOCKDIM_X]
           + s_Data[threadIdx.z][threadIdx.y - 1][threadIdx.x + i * HES_BLOCKDIM_X]
           - s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * HES_BLOCKDIM_X] * 2;
        yz = s_Data[threadIdx.z + 1][threadIdx.y + 1][threadIdx.x + i * HES_BLOCKDIM_X]
           + s_Data[threadIdx.z - 1][threadIdx.y - 1][threadIdx.x + i * HES_BLOCKDIM_X]
           - s_Data[threadIdx.z + 1][threadIdx.y - 1][threadIdx.x + i * HES_BLOCKDIM_X]
           - s_Data[threadIdx.z - 1][threadIdx.y + 1][threadIdx.x + i * HES_BLOCKDIM_X];
        zz = s_Data[threadIdx.z + 1][threadIdx.y][threadIdx.x + i * HES_BLOCKDIM_X]
           + s_Data[threadIdx.z - 1][threadIdx.y][threadIdx.x + i * HES_BLOCKDIM_X]
           - s_Data[threadIdx.z][threadIdx.y][threadIdx.x + i * HES_BLOCKDIM_X] * 2;

        xy *= 0.25f; 
        xz *= 0.25f; 
        yz *= 0.25f;

        d_hessian_pd[i * HES_BLOCKDIM_X] = (xx < 0 && (xx * yy - xy * xy) < 0 && (xx * yy * zz + 2 * xy * yz * xz - xx * yz * yz - yy * xz * xz - zz * xy * xy) < 0) ? 1 : 0;
    }
}