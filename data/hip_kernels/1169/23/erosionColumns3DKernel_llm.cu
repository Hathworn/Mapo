#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void erosionColumns3DKernel(unsigned short *d_dst, unsigned short *d_src, int w, int h, int d, int kernel_radius) {
    // Use shared memory with an optimized layout for coalesced access
    __shared__ unsigned short smem[ER_COLUMNS_BLOCKDIM_Z][ER_COLUMNS_BLOCKDIM_X][(ER_COLUMNS_RESULT_STEPS + 2 * ER_COLUMNS_HALO_STEPS) * ER_COLUMNS_BLOCKDIM_Y];

    unsigned short *smem_thread = smem[threadIdx.z][threadIdx.x];

    // Calculate base indices
    const int baseX = blockIdx.x * ER_COLUMNS_BLOCKDIM_X + threadIdx.x;
    const int baseY = (blockIdx.y * ER_COLUMNS_RESULT_STEPS - ER_COLUMNS_HALO_STEPS) * ER_COLUMNS_BLOCKDIM_Y + threadIdx.y;
    const int baseZ = blockIdx.z * ER_COLUMNS_BLOCKDIM_Z + threadIdx.z;

    d_src += (baseZ * h + baseY) * w + baseX;
    d_dst += (baseZ * h + baseY) * w + baseX;

    // Load main data into shared memory
    #pragma unroll
    for (int i = ER_COLUMNS_HALO_STEPS; i < ER_COLUMNS_HALO_STEPS + ER_COLUMNS_RESULT_STEPS; i++) {
        smem_thread[threadIdx.y + i * ER_COLUMNS_BLOCKDIM_Y] = (baseY + i * ER_COLUMNS_BLOCKDIM_Y < h) ? d_src[i * ER_COLUMNS_BLOCKDIM_Y * w] : 0;
    }

    // Load upper halo
    #pragma unroll
    for (int i = 0; i < ER_COLUMNS_HALO_STEPS; i++) {
        smem_thread[threadIdx.y + i * ER_COLUMNS_BLOCKDIM_Y] = (baseY + i * ER_COLUMNS_BLOCKDIM_Y >= 0) ? d_src[i * ER_COLUMNS_BLOCKDIM_Y * w] : 0;
    }

    // Load lower halo
    #pragma unroll
    for (int i = ER_COLUMNS_HALO_STEPS + ER_COLUMNS_RESULT_STEPS; i < ER_COLUMNS_HALO_STEPS + ER_COLUMNS_RESULT_STEPS + ER_COLUMNS_HALO_STEPS; i++) {
        smem_thread[threadIdx.y + i * ER_COLUMNS_BLOCKDIM_Y] = (baseY + i * ER_COLUMNS_BLOCKDIM_Y < h) ? d_src[i * ER_COLUMNS_BLOCKDIM_Y * w] : 0;
    }

    // Synchronize threads before computation
    __syncthreads();

    // Compute and store results
    #pragma unroll
    for (int i = ER_COLUMNS_HALO_STEPS; i < ER_COLUMNS_HALO_STEPS + ER_COLUMNS_RESULT_STEPS; i++) {
        unsigned short *smem_kern = &smem_thread[threadIdx.y + i * ER_COLUMNS_BLOCKDIM_Y - kernel_radius];
        unsigned short val = smem_kern[0];

        #pragma unroll
        for (int j = 1; j <= 2 * kernel_radius; j++) {
            val = min(val, smem_kern[j]);
        }
        d_dst[i * ER_COLUMNS_BLOCKDIM_Y * w] = val;
    }
}