#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gradientColumnsKernel( float *d_Dst, float *d_Src, int imageW, int imageH, int imageD )
{
    __shared__ float s_Data[COLUMNS_GRAD_BLOCKDIM_Z][COLUMNS_GRAD_BLOCKDIM_X][(COLUMNS_GRAD_RESULT_STEPS + 2 * COLUMNS_GRAD_HALO_STEPS) * COLUMNS_GRAD_BLOCKDIM_Y + 1];

    // Offset calculations
    const int baseX = blockIdx.x * COLUMNS_GRAD_BLOCKDIM_X + threadIdx.x;
    const int baseY = (blockIdx.y * COLUMNS_GRAD_RESULT_STEPS - COLUMNS_GRAD_HALO_STEPS) * COLUMNS_GRAD_BLOCKDIM_Y + threadIdx.y;
    const int baseZ = blockIdx.z * COLUMNS_GRAD_BLOCKDIM_Z + threadIdx.z;
    d_Src += (baseZ * imageH + baseY) * imageW + baseX;
    d_Dst += (baseZ * imageH + baseY) * imageW + baseX;

    // Load main data into shared memory
    #pragma unroll
    for (int i = threadIdx.y; i < COLUMNS_GRAD_BLOCKDIM_Y * (COLUMNS_GRAD_RESULT_STEPS + 2 * COLUMNS_GRAD_HALO_STEPS); i += COLUMNS_GRAD_BLOCKDIM_Y) {
        int idx = i + COLUMNS_GRAD_HALO_STEPS * COLUMNS_GRAD_BLOCKDIM_Y;
        s_Data[threadIdx.z][threadIdx.x][idx] = (baseY + i >= 0 && baseY + i < imageH) ? d_Src[i * imageW] : 0.0f;
    }
    
    // Compute and store results
    __syncthreads();
    #pragma unroll
    for (int i = COLUMNS_GRAD_HALO_STEPS; i < COLUMNS_GRAD_HALO_STEPS + COLUMNS_GRAD_RESULT_STEPS; i++) {
        int idx = threadIdx.y + i * COLUMNS_GRAD_BLOCKDIM_Y;
        float sum = 0.5f * (s_Data[threadIdx.z][threadIdx.x][idx + 1] - s_Data[threadIdx.z][threadIdx.x][idx - 1]);
        d_Dst[i * COLUMNS_GRAD_BLOCKDIM_Y * imageW] = sum;
    }
}