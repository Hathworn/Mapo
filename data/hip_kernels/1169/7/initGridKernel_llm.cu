#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initGridKernel (float *d_grid, int axis, int w, int h, int d) {
    // Use shared memory to reduce global memory accesses
    __shared__ float shared_grid[IG_BLOCKDIM_X * IG_BLOCKDIM_Y * IG_BLOCKDIM_Z];

    const int baseX = blockIdx.x * IG_BLOCKDIM_X + threadIdx.x;
    const int baseY = blockIdx.y * IG_BLOCKDIM_Y + threadIdx.y;
    const int baseZ = blockIdx.z * IG_BLOCKDIM_Z + threadIdx.z;
    const int idx = (baseZ * h + baseY) * w + baseX;
    const int localIdx = threadIdx.z * IG_BLOCKDIM_X * IG_BLOCKDIM_Y + threadIdx.y * IG_BLOCKDIM_X + threadIdx.x;

    // Store values in shared memory to reduce global writes 
    if (axis == 0) {
        shared_grid[localIdx] = (float)baseX;
    } else if (axis == 1) {
        shared_grid[localIdx] = (float)baseY;
    } else {
        shared_grid[localIdx] = (float)baseZ;
    }
    
    __syncthreads();
    
    // Write back to global memory
    d_grid[idx] = shared_grid[localIdx];
}