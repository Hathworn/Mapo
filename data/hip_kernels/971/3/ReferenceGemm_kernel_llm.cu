#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ReferenceGemm_kernel(int M, int N, int K, float alpha, float const *A, int lda, float const *B, int ldb, float beta, float *C, int ldc) {
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Changed index variable names
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Changed index variable names

    if (row < M && col < N) { // Updated with more intuitive variable names
        float accumulator = 0.0f;

        for (int k = 0; k < K; ++k) {
            accumulator += A[row + k * lda] * B[k + col * ldb]; // Optimized indexing order
        }

        C[row + col * ldc] = alpha * accumulator + beta * C[row + col * ldc];
    }
}