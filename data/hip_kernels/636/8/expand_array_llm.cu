#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void expand_array(unsigned char *d_in, unsigned char *d_out)
{
    uint32_t offset = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned char *input = d_in + offset * 5;   // No need to multiply by sizeof(unsigned char)
    unsigned char *output = d_out + offset * 6; // No need to multiply by sizeof(unsigned char)

    // Use local variables to store intermediate results for better readability and potential optimizations
    unsigned char in0 = input[0];
    unsigned char in1 = input[1];
    unsigned char in2 = input[2];

    output[0] = in0 >> 4;
    output[1] = (in0 << 4) | (in1 >> 4);
    output[2] = (in1 << 4) | (in2 >> 4);
    output[3] = in2 & 0xf;
    output[4] = input[3];
    output[5] = input[4];
}