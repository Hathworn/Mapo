#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void map_index_to_prefix(uint8_t *hashes, uint32_t *sort_indices, uint32_t *comb_count, uint32_t *comb_sum, uint32_t *comb_prefix, uint32_t r, uint32_t size)
{
    uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;
    if (t_index < size) {
        uint32_t index = sort_indices[t_index];
        unsigned char *hash = hashes + index * 30 + r * 3;  // Simplified pointer arithmetic
        uint32_t key = (hash[0] << 16) | (hash[1] << 8) | hash[2];

        uint32_t count = comb_count[key];
        uint32_t sum = comb_sum[key];
        for (uint32_t i = sum - count; i < sum; i++) {  // Use uint32_t for loop index
            comb_prefix[i] = key;
        }
    }
}