#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void final_mark_starts( uint8_t *hashes, uint32_t *sort_indices, uint32_t *off_map, uint32_t r, uint32_t hash_count)
{
    // Calculate the global thread index
    uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;
    if(t_index < hash_count) {
        uint32_t t_prev_index = (t_index + hash_count - 1) % hash_count; // Better wrap around
        uint32_t index = sort_indices[t_index];
        uint32_t prev_index = sort_indices[t_prev_index];

        // Use pointers to access hashes more efficiently
        unsigned char* hash = hashes + index * 30 + r * 3;
        unsigned char* prev_hash = hashes + prev_index * 30 + r * 3;

        // Consolidate key and prev_key computation in one step for efficiency
        uint64_t key = ((uint64_t)hash[0] << 40) | ((uint64_t)hash[1] << 32) | (hash[2] << 24) | (hash[3] << 16) | (hash[4] << 8) | hash[5];
        uint64_t prev_key = ((uint64_t)prev_hash[0] << 40) | ((uint64_t)prev_hash[1] << 32) | (prev_hash[2] << 24) | (prev_hash[3] << 16) | (prev_hash[4] << 8) | prev_hash[5];

        // Use ^ to compare keys, update off_map if necessary
        if((key ^ prev_key) != 0) {
            off_map[t_index] = 1;
        }
    }
}