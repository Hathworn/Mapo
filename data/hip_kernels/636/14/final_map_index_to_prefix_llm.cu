#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void final_map_index_to_prefix(uint8_t *hashes, uint32_t *sort_indices, uint32_t *off_map, uint32_t *comb_count, uint32_t *comb_sum, uint32_t *comb_prefix, uint32_t r, uint32_t size)
{
    uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;
    if(t_index < size) {
        uint32_t index = sort_indices[t_index];
        // Simplify hash address calculation
        const unsigned char* hash = &hashes[index * 30 + r * 3];

        // Use 64-bit operations directly for key calculation
        uint64_t key = ((uint64_t)hash[0] << 40) | ((uint64_t)hash[1] << 32) | ((uint64_t)hash[2] << 24) 
                     | ((uint64_t)hash[3] << 16) | ((uint64_t)hash[4] << 8) | (uint64_t)hash[5];

        uint32_t lookup_idx = off_map[t_index];
        uint64_t count = comb_count[lookup_idx];
        uint64_t sum = comb_sum[lookup_idx];
        
        // Unroll loop for efficiency
        for(uint64_t i = sum - count; i < sum; ++i) {
            comb_prefix[i] = lookup_idx;
        }
    }
}