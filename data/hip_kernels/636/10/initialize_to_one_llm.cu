#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initialize_to_one(uint32_t *reduction, uint32_t size)
{
    // Use blockIdx.x * blockDim.x + threadIdx.x for index calculation
    uint32_t t_index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we do not overflow the array bounds
    if (t_index < size) {
        reduction[t_index] = 1;
    }
}