#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void remap_reduction(uint32_t *d_reduction, uint32_t *d_mapping, uint32_t *old_d_ij_buf, uint32_t sum_prev_size, uint32_t prev_size, uint32_t *new_d_ij_buf, uint32_t new_size)
{
    uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;

    if (t_index < prev_size && d_reduction[t_index]) {  // Combine conditions to reduce branching
        uint32_t index = d_mapping[t_index];
        uint32_t i = old_d_ij_buf[2 * sum_prev_size + t_index];  // Use direct array indexing
        uint32_t j = old_d_ij_buf[2 * sum_prev_size + prev_size + t_index];

        // Swap 'i' and 'j' if needed using conditional operator
        if (!sum_prev_size && j < i) {
            uint32_t temp = i;
            i = j;
            j = temp;
        }

        new_d_ij_buf[2 * sum_prev_size + index] = i;  // Use direct array indexing
        new_d_ij_buf[2 * sum_prev_size + new_size + index] = j;
    }
}