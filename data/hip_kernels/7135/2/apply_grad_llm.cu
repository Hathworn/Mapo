#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void apply_grad(float *output, float *grad, const int N)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x; // Improved stride calculation for threads

    for (int idx = pos; idx < N; idx += stride) { // More efficient loop based on stride
        output[idx] += dt * grad[idx];
    }
}