#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float step_function(float v) // Sigmoid function::Activation Function
{
    return 1 / (1 + exp(-v));
}

__global__ void apply_step_function(float *input, float *output, const int N)
{
    // Calculate the global index of the thread
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the index is within bounds
    if (idx < N) {
        output[idx] = step_function(input[idx]); // Apply step function to each element
    }
}