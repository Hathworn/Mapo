#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void makeError(float *err, float *output, unsigned int Y, const int N)
{
    // Compute global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use consistent naming for total threads
    const int totalThreads = blockDim.x * gridDim.x;

    // Calculate stride for each thread and perform loop with stride
    for (int i = idx; i < N; i += totalThreads) {
        err[i] = ((Y == i ? 1.0f : 0.0f) - output[i]);
    }
}