#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __hashmult2(int nrows, int nfeats, int ncols, int brows1, int brows2, float *A, float *Bdata, int *Bir, int *Bjc, float *C, int transpose) {
    // Calculate row and column using thread and block indices for improved parallelization
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if row and column are within bounds
    if (row < nrows && col < ncols) {
        float sum = 0.0f;
        for (int idx = Bjc[col]; idx < Bjc[col + 1]; ++idx) {
            int bRow = Bir[idx];
            sum += A[row * nfeats + bRow] * Bdata[idx];
        }
        C[row * ncols + col] = sum;
    }
}