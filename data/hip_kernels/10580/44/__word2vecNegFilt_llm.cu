#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __word2vecNegFilt(int nrows, int ncols, int nwords, int *WA, int *WB, float *A, float *B, float lrate, float vexp) {
    // Calculate unique index for each thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread is within bounds
    if (idx < nrows * ncols) {
        int row = idx / ncols;
        int col = idx % ncols;

        // Load values to registers to minimize global memory access
        float a_val = A[row * ncols + col];
        int wa_val = WA[row];
        int wb_val = WB[col];

        // Perform computations
        float grad = lrate * a_val * powf((float)(wa_val + wb_val), vexp);

        // Update weight matrices A and B
        A[row * ncols + col] = a_val - grad;
        B[wa_val * ncols + col] += grad;
    }
}