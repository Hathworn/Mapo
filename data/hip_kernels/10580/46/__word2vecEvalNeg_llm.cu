#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __word2vecEvalNeg(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float *Retval) {
    // Calculate unique thread identifier
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < nrows) {
        float sum = 0.0f;
        // Optimize memory access pattern by using shared memory
        __shared__ float sharedA[256];  
        __shared__ float sharedB[256];

        for (int col = 0; col < ncols; col += 256) {
            int idx = col + threadIdx.x;
            if (idx < ncols) {
                sharedA[threadIdx.x] = A[row * ncols + idx];
                sharedB[threadIdx.x] = B[idx];
            }
            __syncthreads();

            for (int i = 0; i < 256; ++i) {
                if (col + i < ncols) {
                    sum += sharedA[i] * sharedB[i];
                }
            }
            __syncthreads();
        }
        Retval[row] = sum;
    }
}