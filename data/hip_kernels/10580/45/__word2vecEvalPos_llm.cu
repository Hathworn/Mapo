#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __word2vecEvalPos(int nrows, int ncols, int *W, int *LB, int *UB, float *A, float *B, float *Retval) {
    // Calculate the unique thread index within the grid
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure that the thread only processes valid data
    if (row < nrows) {
        float temp = 0.0f; // Initialize temporary accumulator variable
        int start = LB[row]; // Get lower bound for the current row
        int end = UB[row]; // Get upper bound for the current row
        for (int col = start; col <= end; ++col) { 
            temp += A[row * ncols + col] * B[col]; // Accumulate partial result
        }
        Retval[row] = temp; // Store result
    }
}