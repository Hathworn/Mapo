#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __findBoundaries(long long *keys, int *jc, int n, int njc, int shift) {
    __shared__ int dbuff[1024];
    int tid = threadIdx.x + blockDim.x * threadIdx.y;
    int imin = ((int)(32 * ((((long long)n) * blockIdx.x) / (gridDim.x * 32))));
    int imax = min(n, ((int)(32 * ((((long long)n) * (blockIdx.x + 1)) / (gridDim.x * 32) + 1))));

    // Initialize first index
    if (tid == 0 && blockIdx.x == 0) {
        jc[0] = 0;
    }
    
    __syncthreads(); // Make sure jc[0] is set before proceeding

    int lasti = 0x7fffffff;
    
    for (int i = imin; i <= imax; i += blockDim.x * blockDim.y) {
        int iv = njc;
        if (i + tid < imax) {
            iv = (int)(keys[i + tid] >> shift);
            dbuff[tid] = iv; // Load values into shared memory
        }
        
        __syncthreads(); // Synchronize to ensure all threads have written to shared memory

        if (i + tid < imax || i + tid == n) {
            if (tid > 0) lasti = dbuff[tid - 1]; // Use shared memory values
            if (iv > lasti) {
                for (int j = lasti + 1; j <= iv; j++) {
                    jc[j] = i + tid; // Update boundary
                }
            }
            if (tid == 0) {
                lasti = dbuff[blockDim.x * blockDim.y - 1]; // Update last value efficiently
            }
        }
        __syncthreads(); // Synchronize before next iteration
    }
}