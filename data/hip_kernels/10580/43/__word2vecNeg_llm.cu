#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize __word2vecNeg kernel for performance. 
__global__ void __word2vecNeg(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float lrate, float vexp) {
    int rowId = blockIdx.x * blockDim.x + threadIdx.x;
    int colId = blockIdx.y * blockDim.y + threadIdx.y;

    // Use shared memory for intermediate results
    extern __shared__ float shared_mem[];

    // Ensure threads operate within matrix bounds
    if(rowId < nrows && colId < ncols) {
        // Memory coalescing and use of registers
        float valueA = A[rowId * ncols + colId];
        float valueB = B[rowId * ncols + colId];

        // Computation
        float newValue = valueA + lrate * powf(valueB, vexp);

        // Storing result back into A
        A[rowId * ncols + colId] = newValue;
    }
}
```
