#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __veccmp(int *a, int *b, int *d) {
    int xa = *a;
    int xb = *b;
    int xc = 0;
    int xd = 0;

    // Use a loop to reduce code repetition
    for (int i = 0; i < 4; i++) {
        asm("vset4.s32.s32.ne" "%0, %1.b%d%d%d%d, %2, %3;" : "=r" (xd) : "r" (xa), "r" (xb), "r" (xc), "i" (i));
        *d++ = xd;  // Increment pointer and store result
    }
}