#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __word2vecPos(int nrows, int ncols, int *W, int *LB, int *UB, float *A, float *B, float lrate) {
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (index >= nrows) return;

    // Load relevant row
    int row = W[index];

    // Iterate over range defined by LB and UB
    for (int col = LB[index]; col < UB[index]; col++) {
        // Compute and update A and B matrices using lrate
        float temp = A[row * ncols + col] * B[col];
        A[row * ncols + col] += lrate * temp;
        B[col] -= lrate * temp;
    }
}