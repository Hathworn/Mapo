#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __hammingdists(int *a, int *b, int *w, int *op, int *ow, int n) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a loop to handle multiple elements per thread
    for (int i = idx; i < n; i += gridDim.x * blockDim.x) {
        // Example: Compute hamming distance between a[i] and b[i]
        int x = a[i] ^ b[i];
        int dist = 0;

        // Count the number of set bits
        for (; x; x >>= 1)
            dist += x & 1;

        // Store result to output arrays
        op[i] = dist;
        ow[i] = w[i];
    }
}