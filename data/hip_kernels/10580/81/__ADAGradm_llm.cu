#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __ADAGradm(int nrows, int ncols, float *mm, float *um, float *ssq, float *momentum, float mu, float *mask, int maskr, float nw, float *ve, int nve, float *ts, int nts, float *lr, int nlr, float langevin, float eps, int doupdate, hiprandState *rstates) {
    int ithread = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
    int nthreads = blockDim.x * gridDim.x * gridDim.y;
    int i, irow, icol;
    float mmval, umval, sqrtss, sqrtnewss, veval, tsval, lrval, denom, grad;
    
    // Precompute constants outside the loop
    float sqrtnw = sqrtf(nw);
    float sqrt1mnw = sqrtf(1-nw);
    float sqrteps = sqrtf(eps);
    
    hiprandState *prstate = &rstates[ithread];
    
    for (i = ithread; i < nrows * ncols; i += nthreads) {
        icol = i / nrows;
        irow = i - icol * nrows;
        umval = um[i];
        sqrtss = ssq[i];
        
        // Optimized newss calculation using hypotf directly
        sqrtnewss = hypotf(sqrtnw * umval, sqrt1mnw * sqrtss);
        ssq[i] = sqrtnewss;
        
        if (doupdate) {
            mmval = mm[i];
            
            // Use ternary operator for optimized selection
            veval = (nve > 1) ? ve[irow] : ve[0];
            tsval = (nts > 1) ? ts[irow] : ts[0];
            lrval = (nlr > 1) ? lr[irow] : lr[0];
            
            sqrtnewss = hypotf(sqrtnewss, sqrteps);
            denom = (veval == 0.5f) ? sqrtnewss : powf(sqrtnewss, veval * 2);
            grad = umval / denom;
            
            // Add Langevin noise if applicable
            if (langevin > 0) grad += hiprand_normal(prstate) * langevin;
            
            // Optimized gradient update
            grad = grad * lrval * tsval + mu * momentum[i];
            momentum[i] = grad; // Save it
            mmval += grad; // Add the new gradient
            
            // Apply mask if necessary
            if (maskr > 0) {
                mmval *= (maskr > 1) ? mask[i] : mask[icol];
            }
            mm[i] = mmval;
        }
    }
}