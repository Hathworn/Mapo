#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __veccmp(int *a, int *b, int *d) {
    // Get the unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate comparison only if index is in array bound
    if (idx < blockDim.x * gridDim.x) {
        d[idx] = a[idx] > b[idx] ? 1 : (a[idx] < b[idx] ? -1 : 0); // Simplified ternary operation
    }
}