#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __pairmult2(int nrows, int bncols, int brows1, int brows2, float *A, int lda, float *A2, int lda2, float *Bdata, int *Bir, int *Bjc, int broff, int bcoff, float *C, int ldc, int transpose) {
    // Calculate the global row and column index
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary Check
    if (row < nrows && col < bncols) {
        // Initialize the result accumulation variable
        float sum = 0.0f;
        
        // Main computation loop
        for (int idx = Bjc[col + bcoff]; idx < Bjc[col + bcoff + 1]; ++idx) {
            int b_row = Bir[idx];
            if (b_row >= broff) {
                b_row -= broff;
                if (b_row < brows1 || (b_row >= brows1 + brows2 && b_row < nrows)) {
                    if (transpose) {
                        sum += A[b_row * lda + row] * Bdata[idx];
                    } else {
                        sum += A[row * lda + b_row] * Bdata[idx];
                    }
                }
            }
        }
        
        // Write the result
        C[row * ldc + col] = sum;
    }
}