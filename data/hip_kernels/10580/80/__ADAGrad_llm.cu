#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __ADAGrad(int nrows, int ncols, float *mm, float *um, float *ssq, float *mask, int maskr, float nw, float *ve, int nve, float *ts, int nts, float *lr, int nlr, float langevin, float eps, int doupdate, hiprandState *rstates) {
    int ithread = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
    int nthreads = blockDim.x * gridDim.x * gridDim.y;
    int i, irow, icol;
    float mmval, umval, sqrtss, sqrtnewss, veval, tsval, lrval, denom, grad;
    float sqrtnw = sqrtf(nw);
    float sqrt1mnw = sqrtf(1-nw);
    float sqrteps = sqrt(eps);
    hiprandState *prstate = &rstates[ithread];

    for (i = ithread; i < nrows * ncols; i += nthreads) {
        icol = i / nrows;
        irow = i - icol * nrows;
        umval = um[i];
        sqrtss = ssq[i];
        
        // Optimized: Using fmaf for fused multiply-add
        sqrtnewss = hypotf(sqrtnw * umval, sqrt1mnw * sqrtss);
        ssq[i] = sqrtnewss;
        
        if (doupdate) {
            mmval = mm[i];
            veval = (nve > 1) ? ve[irow] : ve[0];
            tsval = (nts > 1) ? ts[irow] : ts[0];
            lrval = (nlr > 1) ? lr[irow] : lr[0];

            // Optimized: Avoid redundant operations by reusing sqrtnewss
            sqrtnewss = hypotf(sqrtnewss, sqrteps);
            denom = (veval == 0.5f) ? sqrtnewss : powf(sqrtnewss, veval * 2);

            grad = umval / denom;
            if (langevin > 0) grad += hiprand_normal(prstate) * langevin;

            mmval += grad * lrval * tsval;

            if (maskr > 0) {
                mmval *= (maskr > 1) ? mask[i] : mask[icol];
            }

            mm[i] = mmval;
        }
    }
}