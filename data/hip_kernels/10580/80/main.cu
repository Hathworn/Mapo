#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "__ADAGrad.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    int nrows = 1;
int ncols = 1;
float *mm = NULL;
hipMalloc(&mm, XSIZE*YSIZE);
float *um = NULL;
hipMalloc(&um, XSIZE*YSIZE);
float *ssq = NULL;
hipMalloc(&ssq, XSIZE*YSIZE);
float *mask = NULL;
hipMalloc(&mask, XSIZE*YSIZE);
int maskr = 1;
float nw = 1;
float *ve = NULL;
hipMalloc(&ve, XSIZE*YSIZE);
int nve = 1;
float *ts = NULL;
hipMalloc(&ts, XSIZE*YSIZE);
int nts = 1;
float *lr = NULL;
hipMalloc(&lr, XSIZE*YSIZE);
int nlr = 1;
float langevin = 1;
float eps = 1;
int doupdate = 1;
hiprandState *rstates = NULL;
hipMalloc(&rstates, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
__ADAGrad<<<gridBlock, threadBlock>>>(nrows,ncols,mm,um,ssq,mask,maskr,nw,ve,nve,ts,nts,lr,nlr,langevin,eps,doupdate,rstates);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
__ADAGrad<<<gridBlock, threadBlock>>>(nrows,ncols,mm,um,ssq,mask,maskr,nw,ve,nve,ts,nts,lr,nlr,langevin,eps,doupdate,rstates);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
__ADAGrad<<<gridBlock, threadBlock>>>(nrows,ncols,mm,um,ssq,mask,maskr,nw,ve,nve,ts,nts,lr,nlr,langevin,eps,doupdate,rstates);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}