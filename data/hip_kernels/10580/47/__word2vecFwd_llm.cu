#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __word2vecFwd(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float *C) {
    // Use shared memory to reduce global memory accesses
    extern __shared__ float sA[], sB[];

    // Calculate thread indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory
    if (row < nrows && threadIdx.x < ncols) {
        sA[threadIdx.y * ncols + threadIdx.x] = A[row * ncols + threadIdx.x];
        sB[threadIdx.y * ncols + threadIdx.x] = B[row * ncols + threadIdx.x];
    }
    __syncthreads();

    // Compute using shared memory
    if (row < nrows && col < ncols) {
        C[row * ncols + col] = sA[threadIdx.y * ncols + col] + sB[threadIdx.y * ncols + col];
    }
}