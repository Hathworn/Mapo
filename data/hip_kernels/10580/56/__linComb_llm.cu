#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void __linComb(float *X, float wx, float *Y, float wy, float *Z, int len) {
// Calculate global thread ID
int idx = blockIdx.x * blockDim.x + threadIdx.x;
int stride = blockDim.x * gridDim.x;

// Loop through the array using a stride
for (int i = idx; i < len; i += stride) {
    Z[i] = X[i]*wx + Y[i]*wy; // Compute the linear combination
}
}