#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void __word2vecBwd(int nrows, int ncols, int *WA, int *WB, float *A, float *B, float *C, float lrate) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;  // Compute row based on block and thread indices
    if (row < nrows) {  // Check if row index is within bounds
        int w = WA[row];  // Load WA and WB only once
        int w2 = WB[row];
        float *a = &A[w * ncols];  // Compute offset for A and B
        float *b = &B[w2 * ncols];
        float *c = &C[row * ncols];
        for (int col = 0; col < ncols; ++col) {  // Loop through columns
            float grad = c[col] * lrate;  // Compute gradient multiplied by the learning rate
            atomicAdd(&a[col], grad);  // Atomic addition to prevent data races
            atomicAdd(&b[col], grad);  // Atomic addition to prevent data races
        }
    }
}