#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void __ADAGradn(int nrows, int ncols, float *mm, float *um, float *ssq, float *momentum, float mu, float *mask, int maskr, float nw, float *ve, int nve, float *ts, int nts, float *lr, int nlr, float langevin, float eps, int doupdate, hiprandState *rstates) {
    int ithread = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
    int nthreads = blockDim.x * gridDim.x * gridDim.y;

    float sqrtnw = sqrtf(nw);
    float sqrt1mnw = sqrtf(1-nw);
    float sqrteps = sqrt(eps);
    hiprandState *prstate = &rstates[ithread];

    for (int i = ithread; i < nrows * ncols; i += nthreads) {
        int icol = i / nrows;
        int irow = i - icol * nrows;
        
        // Use shared memory for frequently accessed data
        __shared__ float shared_ve, shared_ts, shared_lr;

        if (threadIdx.x == 0) {
            shared_ve = (nve > 1) ? ve[irow] : ve[0];
            shared_ts = (nts > 1) ? ts[irow] : ts[0];
            shared_lr = (nlr > 1) ? lr[irow] : lr[0];
        }
        __syncthreads();

        float umval = um[i];
        float sqrtss = ssq[i];
        
        float sqrtnewss = hypotf(sqrtnw * umval, sqrt1mnw * sqrtss);
        ssq[i] = sqrtnewss;
        
        if (doupdate) {
            float mmval = mm[i];
            sqrtnewss = hypotf(sqrtnewss, sqrteps);
            float denom = (shared_ve == 0.5f) ? sqrtnewss : powf(sqrtnewss, shared_ve * 2);

            float grad = umval / denom;
            if (langevin > 0) grad += hiprand_normal(prstate) * langevin;
            grad *= shared_lr * shared_ts;                 // Normal gradient

            float oldmom = momentum[i];                     // Momentum
            float newmom = grad + mu * oldmom;              // Compute new momentum
            momentum[i] = newmom;                           // Save new momentum
            
            mmval += newmom + mu * (newmom - oldmom);       // Update mmval
            
            if (maskr > 0) {
                if (maskr > 1) {
                    mmval *= mask[i];
                } else {
                    mmval *= mask[icol];
                }
            }
            mm[i] = mmval;
        }
    }
}