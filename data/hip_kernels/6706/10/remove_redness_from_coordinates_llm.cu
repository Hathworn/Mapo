#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void remove_redness_from_coordinates(const unsigned int* d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, int num_coordinates, int num_pixels_y, int num_pixels_x, int template_half_height, int template_half_width) 
{
    int nx = num_pixels_x;
    int ny = num_pixels_y;
    int global_index_1d = (blockIdx.x * blockDim.x) + threadIdx.x;
    int imgSize = nx * ny;

    if (global_index_1d < num_coordinates) 
    {
        unsigned int image_index_1d = d_coordinates[imgSize - global_index_1d - 1];
        ushort2 image_index_2d = make_ushort2(image_index_1d % nx, image_index_1d / nx);

        // Loop unrolling for improved latency hiding
        #pragma unroll
        for (int dy = -template_half_height; dy <= template_half_height; dy++) 
        {
            int y = image_index_2d.y + dy;
            if (y < 0 || y >= ny) continue; // Skip if out of bounds
            #pragma unroll
            for (int dx = -template_half_width; dx <= template_half_width; dx++) 
            {
                int x = image_index_2d.x + dx;
                if (x < 0 || x >= nx) continue; // Skip if out of bounds

                int image_offset_index_1d_clamped = (nx * y) + x;
                unsigned char g_value = d_g[image_offset_index_1d_clamped];
                unsigned char b_value = d_b[image_offset_index_1d_clamped];
                unsigned int gb_average = (g_value + b_value) / 2;

                d_r_output[image_offset_index_1d_clamped] = (unsigned char)gb_average;
            }
        }
    }
}