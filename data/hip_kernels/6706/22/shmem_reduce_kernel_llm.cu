#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shmem_reduce_kernel(float * d_out, const float * d_in)
{
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // Load shared mem from global mem
    sdata[tid] = d_in[myId];
    __syncthreads();

    // Use loop unrolling for reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}