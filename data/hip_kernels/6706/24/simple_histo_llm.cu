#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    // Compute global ID
    int myId = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Read item only if the thread is within data bounds
    if(myId < BIN_COUNT) {
        int myItem = d_in[myId];
        
        // Compute bin only if the index is valid
        int myBin = myItem % BIN_COUNT;
        
        // Use atomic operation safely for valid bins
        atomicAdd(&(d_bins[myBin]), 1);
    }
}