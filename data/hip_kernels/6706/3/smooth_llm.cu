#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth(float * v_new, const float * v) {
    int myIdx = threadIdx.x + blockIdx.x * blockDim.x;  // Corrected index calculation
    int numElements = gridDim.x * blockDim.x;  // Total number of elements

    // Guard against accessing out-of-bound elements
    if (myIdx < numElements) {
        int myLeftIdx = max(0, myIdx - 1);  // Use max to handle edge case
        int myRightIdx = min(numElements - 1, myIdx + 1);  // Use min to handle edge case

        float myElt = v[myIdx];
        float myLeftElt = v[myLeftIdx];
        float myRightElt = v[myRightIdx];
        v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
    }
}