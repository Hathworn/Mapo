#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
    unsigned int lane = threadIdx.x % 32; // Calculate lane index within a warp

    // Perform warp-level reduction using shuffle operations
    for (int offset = 16; offset > 0; offset /= 2) {
        p += __shfl_down_sync(0xffffffff, p, offset);
    }

    if (lane == 0) s[threadIdx.x / 32] = p; // Write reduced value of this warp to shared memory

    __syncthreads(); // Ensure all warps have written their results

    // Use the first warp to reduce values across warps
    if (threadIdx.x < 32) {
        unsigned int val = (threadIdx.x < (blockDim.x / 32)) ? s[threadIdx.x] : 0;
        for (int offset = 16; offset > 0; offset /= 2) {
            val += __shfl_down_sync(0xffffffff, val, offset);
        }
        if (threadIdx.x == 0) s[0] = val; // Store the final result in s[0]
    }

    __syncthreads(); // Ensure the final result is available for all threads

    return s[0]; // Return the reduced value
}

__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    if (t == 0) {
        *d_out_shared = sr;
    }
}