#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // Load input into shared memory, one element per thread
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    // Bitonic sorting network
    for (int stage = 0; stage <= 5; stage++)
    {
        for (int substage = stage; substage >= 0; substage--)
        {
            int pair_distance = 1 << substage;
            int block_distance = 1 << (stage + 1);

            int compare_index = 2 * pair_distance * (tid / (2 * pair_distance)) + (tid % pair_distance);
            int first_index = compare_index + pair_distance;
            int second_index = first_index + pair_distance;

            if ((tid % block_distance) < block_distance / 2)
            {
                if (sdata[first_index] > sdata[second_index])
                {
                    // Swap elements to sort in ascending order
                    float temp = sdata[first_index];
                    sdata[first_index] = sdata[second_index];
                    sdata[second_index] = temp;
                }
            }
            else
            {
                if (sdata[first_index] < sdata[second_index])
                {
                    // Swap elements to sort in descending order
                    float temp = sdata[first_index];
                    sdata[first_index] = sdata[second_index];
                    sdata[second_index] = temp;
                }
            }
            __syncthreads();
        }
    }

    // Write sorted data to global memory
    d_out[tid] = sdata[tid];
}