#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global index
    printf("Hello world! I'm thread %d\n", idx);     // Use global index for unique identification
}