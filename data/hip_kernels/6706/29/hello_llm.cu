#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Each block calculates its unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Print statement optimized to include thread ID
    printf("Hello world! I'm thread %d in block %d\n", tid, blockIdx.x);
}