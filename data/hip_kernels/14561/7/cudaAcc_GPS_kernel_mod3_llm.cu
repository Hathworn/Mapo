#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cudaAcc_GPS_kernel_mod3(int NumDataPoints, float2* FreqData, float* PowerSpectrum)
{
    const int sidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check index within bounds
    if (sidx < NumDataPoints)
    {
        float2 freq = FreqData[sidx]; // Load once, use registers
        float ax = freq.x;
        float ay = freq.y;
        
        // Compute power spectrum using FMA
        PowerSpectrum[sidx] = __fmaf_rn(ax, ax, ay * ay);
    }
}