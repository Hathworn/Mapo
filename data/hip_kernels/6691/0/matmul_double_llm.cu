#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void matmul_double(double* A, double* B , double* C, int M, int N, int K)
{
    // Thread and block indices
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Calculate row and column indices
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    // Allocate shared memory
    __shared__ double SA[TILE_WIDTH][TILE_WIDTH];
    __shared__ double SB[TILE_WIDTH][TILE_WIDTH];

    double Csub = 0;

    for (int i = 0; i < (K + TILE_WIDTH - 1) / TILE_WIDTH; ++i)
    {
        // Load elements into shared memory with boundary checks
        if (row < M && i * TILE_WIDTH + tx < K) {
            SA[ty][tx] = A[row * K + i * TILE_WIDTH + tx];
        } else {
            SA[ty][tx] = 0.0;
        }

        if (col < N && i * TILE_WIDTH + ty < K) {
            SB[ty][tx] = B[(i * TILE_WIDTH + ty) * N + col];
        } else {
            SB[ty][tx] = 0.0;
        }

        __syncthreads();

        // Compute partial product for the tile
        for (int k = 0; k < TILE_WIDTH; ++k) {
            Csub += SA[ty][k] * SB[k][tx];
        }

        __syncthreads();
    }

    // Write result to global memory if within bounds
    if (row < M && col < N) {
        C[row * N + col] = Csub;
    }
}