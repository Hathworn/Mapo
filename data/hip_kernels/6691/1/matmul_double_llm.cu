#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matmul_double(double* A, double* B ,  double* C, int M, int N, int K)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    // Align shared memory for better performance in coalesced access
    __shared__ double SA[TILE_WIDTH][TILE_WIDTH + 1];
    __shared__ double SB[TILE_WIDTH][TILE_WIDTH + 1];

    double Csub = 0;

    // Loop over the A and B tiles required to compute the C element
    for (int i = 0; i < (K - 1) / TILE_WIDTH + 1; ++i)
    {
        // Load A and B tiles into shared memory; ensure boundary checks
        if ((row < M) && (i * TILE_WIDTH + tx < K)) {
            SA[ty][tx] = A[row * K + i * TILE_WIDTH + tx];
        } else {
            SA[ty][tx] = 0;
        }

        if ((col < N) && (i * TILE_WIDTH + ty < K)) {
            SB[tx][ty] = B[col * K + i * TILE_WIDTH + ty];
        } else {
            SB[tx][ty] = 0;
        }

        __syncthreads(); // Synchronize to wait for tile data to be loaded

        // Compute matrix multiplication for the tile
        #pragma unroll // Unroll the loop for better performance
        for (int k = 0; k < TILE_WIDTH; ++k) {
            Csub += SA[ty][k] * SB[tx][k];
        }

        __syncthreads(); // Ensure all threads have finished using shared memory
    }

    // Write the block sub-matrix to global memory
    if ((row < M) && (col < N)) {
        C[row * N + col] = Csub;
    }
}