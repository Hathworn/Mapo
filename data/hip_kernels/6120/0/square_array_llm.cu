#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    // Calculate global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Utilize shared memory to reduce global memory access latency
    __shared__ float shared_a[256]; // Assuming blockDim.x <= 256
    if (idx < N) {
        // Load data into shared memory
        shared_a[threadIdx.x] = a[idx];
        __syncthreads();
        // Perform computation
        shared_a[threadIdx.x] = shared_a[threadIdx.x] * shared_a[threadIdx.x];
        __syncthreads();
        // Store data back to global memory
        a[idx] = shared_a[threadIdx.x];
    }
}