#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMultiply(float* a, float* b, float* c, int n)
{
    // Define shared memory for A and B tiles
    extern __shared__ float shared[];

    // Use block dimensions to calculate column and row
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Declare shared memory pointers for A and B tiles
    float* shared_a = shared;
    float* shared_b = shared + blockDim.x * blockDim.y;

    // Initialize C matrix element to zero
    float cValue = 0.0f;

    // Loop over tiles
    for (int tileIdx = 0; tileIdx < n / blockDim.x; ++tileIdx) {
        
        // Load A and B matrices into shared memory
        shared_a[threadIdx.y * blockDim.x + threadIdx.x] = a[row * n + tileIdx * blockDim.x + threadIdx.x];
        shared_b[threadIdx.y * blockDim.x + threadIdx.x] = b[(tileIdx * blockDim.y + threadIdx.y) * n + col];
        
        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply shared memory elements
        for (int k = 0; k < blockDim.x; ++k) {
            cValue += shared_a[threadIdx.y * blockDim.x + k] * shared_b[k * blockDim.x + threadIdx.x];
        }
        
        // Synchronize before loading the next tile
        __syncthreads();
    }

    // Assign the computed value to the output matrix
    c[row * n + col] = cValue;
}