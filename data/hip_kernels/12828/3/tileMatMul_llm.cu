#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tileMatMul(float* matA, float* matB, float* matC, int aRows, int aCols, int bRows, int bCols, int cRows, int cCols)
{
    // Define row and column values
    int Row = blockIdx.y * TILE_DIM + threadIdx.y;
    int Col = blockIdx.x * TILE_DIM + threadIdx.x;

    // Shared memory arrays
    __shared__ float sharedMatA[TILE_DIM][TILE_DIM];
    __shared__ float sharedMatB[TILE_DIM][TILE_DIM];

    float cResultValue = 0.0f;

    // Calculate tiled matrix multiplication using shared memory
    for(int i = 0; i < (aCols - 1) / TILE_DIM + 1; ++i) {
        // Load elements of matA into shared memory
        if(Row < aRows && i * TILE_DIM + threadIdx.x < aCols) {
            sharedMatA[threadIdx.y][threadIdx.x] = matA[Row * aCols + i * TILE_DIM + threadIdx.x];
        } else {
            sharedMatA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        // Load elements of matB into shared memory
        if(Col < bCols && i * TILE_DIM + threadIdx.y < bRows) { // Fixed to check bounds with bRows
            sharedMatB[threadIdx.y][threadIdx.x] = matB[(i * TILE_DIM + threadIdx.y) * bCols + Col];
        } else {
            sharedMatB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        // Compute partial product for this tile
        for(int j = 0; j < TILE_DIM; ++j) {
            cResultValue += sharedMatA[threadIdx.y][j] * sharedMatB[j][threadIdx.x];
        }
        
        __syncthreads();
    }

    // Write the result to the output matrix
    if(Row < cRows && Col < cCols) {
        matC[Row * cCols + Col] = cResultValue;
    }
}