#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized CUDA kernel
__global__ void vectorSum(int *a, int *b, int *c, int n)
{
    // Compute the global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread ID is within bounds and perform vector addition
    if (id < n)
    {
        c[id] = a[id] + b[id];
    }
}