#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

// Optimized kernel function using blockIdx and blockDim
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if (i < size) { // Check bounds to prevent out-of-range memory access
        c[i] = a[i] + b[i];
    }
}