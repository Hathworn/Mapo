#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum(float *a, float *b, float *c, int n) {
    // Calculate the global index
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure the index is within bounds
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}