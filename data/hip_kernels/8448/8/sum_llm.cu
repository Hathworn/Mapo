#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory for reducing global memory accesses
__global__ void sum(float *a, float *b, float *c, int n) {
  extern __shared__ float shared[];
  int index = blockDim.x * blockIdx.x + threadIdx.x;

  // Check if index is within bounds
  if (index < n) {
    // Load data into shared memory
    shared[threadIdx.x] = a[index];
    shared[threadIdx.x + blockDim.x] = b[index];
    __syncthreads();

    // Perform addition using shared memory
    c[index] = shared[threadIdx.x] + shared[threadIdx.x + blockDim.x];
  }
}