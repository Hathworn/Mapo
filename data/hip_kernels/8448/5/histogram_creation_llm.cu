#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __host__ void print(float *result)
{
    for(int k = 0; k < 3; k++)
    {
        for(int i = 0; i < N; i++)
        {
            for(int j = 0; j < N; j++)
                printf("%f ",result[(i*N + j)*3 + k]);
            printf("\n");
        }
        printf("\n");
    }
}

__global__ void histogram_creation(int *A, int *hist, int no_of_threads) {
    // Calculate global index
    int global_x = blockDim.x * blockIdx.x + threadIdx.x;
    // Declare shared memory for local histogram
    __shared__ int local_hist[N+1];

    // Initialize local histogram using block-wide sync
    for(int i = threadIdx.x; i <= N; i += blockDim.x) {
        local_hist[i] = 0;
    }
    __syncthreads();

    // Populate local histogram using atomic adds
    for(int i = global_x; i < M; i += blockDim.x * gridDim.x) {
        atomicAdd(&local_hist[A[i]], 1);
    }
    __syncthreads();

    // Update global histogram using atomic adds
    for(int i = threadIdx.x; i <= N; i += blockDim.x) {
        atomicAdd(&hist[i], local_hist[i]);
    }
}