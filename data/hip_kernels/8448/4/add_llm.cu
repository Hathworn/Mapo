#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(float *cudaA, float *kernel, float *cudaResult)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int gid = idy * N + idx;

    __shared__ float blockData[BLOCK_SIZE + 2 * BLUR_SIZE][BLOCK_SIZE + 2 * BLUR_SIZE][3];

    // Load the entire blockData, handle boundary conditions
    for (int dy = -BLUR_SIZE; dy <= BLUR_SIZE; dy++) {
        for (int dx = -BLUR_SIZE; dx <= BLUR_SIZE; dx++) {
            int x = idx + dx;
            int y = idy + dy;
            int shared_x = threadIdx.x + BLUR_SIZE + dx;
            int shared_y = threadIdx.y + BLUR_SIZE + dy;
            
            if (x >= 0 && x < N && y >= 0 && y < N) {
                int global_gid = (y * N + x) * 3;
                for (int k = 0; k < 3; k++) {
                    blockData[shared_y][shared_x][k] = cudaA[global_gid + k];
                }
            } else {
                for (int k = 0; k < 3; k++) {
                    blockData[shared_y][shared_x][k] = 0.0f;
                }
            }
        }
    }

    __syncthreads();

    // Apply the kernel
    for (int k = 0; k < 3; k++) {
        float sum = 0.0f;
        for (int i = 0; i <= 2 * BLUR_SIZE; i++) {
            for (int j = 0; j <= 2 * BLUR_SIZE; j++) {
                sum += blockData[threadIdx.y + i][threadIdx.x + j][k] * 
                       kernel[(2 * BLUR_SIZE - i) * (2 * BLUR_SIZE + 1) + (2 * BLUR_SIZE - j)];
            }
        }
        cudaResult[gid * 3 + k] = sum;
    }
}