#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tiled_matrix_multiplication(int *A, int *B, int *C) {

    __shared__ int As[TILE_WIDTH][TILE_WIDTH];
    __shared__ int Bs[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    int res = 0;

    for(int i = 0; i < M/TILE_WIDTH; i++) {
        // Load data into shared memory more efficiently
        int a_index = row * M + (i * TILE_WIDTH + tx);
        int b_index = (i * TILE_WIDTH + ty) * M + col;
        
        As[ty][tx] = A[a_index];
        Bs[ty][tx] = B[b_index];

        __syncthreads();

        // Multiply and accumulate
        for(int j = 0; j < TILE_WIDTH; j++) {
            res += As[ty][j] * Bs[j][tx]; // Fixing the operation to multiplication
        }

        __syncthreads();
    }

    // Store final result
    C[row * M + col] = res;
}