#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    // Use int for idx and pre-compute blockDim.x for efficiency
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float value = a[idx]; // Load value to register
        a[idx] = value * value; // Compute square and store
    }
}