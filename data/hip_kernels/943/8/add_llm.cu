#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(float *loc, float *temp, const int num) {
    // Calculate global thread ID
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid atomic operation in every thread by using shared memory
    __shared__ float tempSum[1024]; // Ensures shared memory space for block

    // Initialize shared memory
    tempSum[threadIdx.x] = (idx < num) ? temp[idx] : 0.0f;
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride && (idx + stride) < num) {
            tempSum[threadIdx.x] += tempSum[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Atomic addition only once per block
    if (threadIdx.x == 0 && idx < num) {
        atomicAdd(loc, tempSum[0]);
    }
}