#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Calculate unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Coalesced access: Use stride pattern for better memory access
    for (int i = tid; i < N; i += gridDim.x * blockDim.x) {
        out[i] = max(0.0f, in[i]);
    }
}