#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Optimize condition check and assignment using ternary operator directly
    if (tid < N) {
        float in_val = indata[tid];
        ingrad[tid] = (in_val > 0.0f) * outgrad[tid]; // Use boolean multiplication for condition
    }
}