#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access by utilizing shared memory
__global__ void add(int *a, int *b, int *c)
{
    extern __shared__ int sharedMem[];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Load data into shared memory
    sharedMem[threadIdx.x] = a[index];
    sharedMem[threadIdx.x + blockDim.x] = b[index];
    __syncthreads();

    // Compute the result using shared memory
    c[index] = sharedMem[threadIdx.x] + sharedMem[threadIdx.x + blockDim.x];
}