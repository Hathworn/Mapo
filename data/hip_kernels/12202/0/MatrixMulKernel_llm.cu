#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// The matrix dimensions
static const int M = 3;
static const int N = 5;
static const int K = 4;
static const int TILE_WIDTH = 2;

// Kernel function for matrix multiplication
__global__ void MatrixMulKernel(int ARows, int ACols, int BRows, int BCols, int CRows, int CCols, unsigned int* A_d, unsigned int* B_d, unsigned int* C_d) {
    // Compute row and column indices
    int Row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int Col = blockIdx.x * TILE_WIDTH + threadIdx.x;

    // Define shared memory for tiles
    __shared__ int As[TILE_WIDTH][TILE_WIDTH];
    __shared__ int Bs[TILE_WIDTH][TILE_WIDTH];

    // Accumulate result for CValue
    int CValue = 0;
    
    for (int k = 0; k < (ACols + TILE_WIDTH - 1)/TILE_WIDTH; ++k) {
        // Load tiles into shared memory
        if (k * TILE_WIDTH + threadIdx.x < ACols && Row < ARows) {
            As[threadIdx.y][threadIdx.x] = A_d[Row * ACols + k * TILE_WIDTH + threadIdx.x];
        } else {
            As[threadIdx.y][threadIdx.x] = 0;
        }

        if (k * TILE_WIDTH + threadIdx.y < BRows && Col < BCols) {
            Bs[threadIdx.y][threadIdx.x] = B_d[(k * TILE_WIDTH + threadIdx.y) * BCols + Col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0;
        }

        __syncthreads();

        // Perform multiplication on the tile
        for (int n = 0; n < TILE_WIDTH; ++n) {
            CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];
        }

        __syncthreads();
    }

    // Write the result to the output matrix
    if (Row < CRows && Col < CCols) {
        C_d[Row * CCols + Col] = CValue;
    }
}