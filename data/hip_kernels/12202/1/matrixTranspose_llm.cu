#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixTranspose(unsigned int* A_d, unsigned int *T_d, int rowCount, int colCount) {

    // Use shared memory for tile-based optimization
    __shared__ unsigned int tile[32][32 + 1]; // Avoid bank conflicts

    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < rowCount && col < colCount) {
        // Load data into shared memory
        tile[threadIdx.y][threadIdx.x] = A_d[row * colCount + col];
    }
    __syncthreads();

    // Transpose block within shared memory
    col = blockIdx.y * blockDim.y + threadIdx.x;
    row = blockIdx.x * blockDim.x + threadIdx.y;

    if (row < colCount && col < rowCount) {
        // Store transposed data from shared memory
        T_d[col * rowCount + row] = tile[threadIdx.x][threadIdx.y];
    }
}