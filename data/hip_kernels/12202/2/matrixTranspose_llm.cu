#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixTranspose(unsigned int* A_d, unsigned int *T_d, int n) {
    // Calculate global row and column index for the thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Transpose the matrix only if indices are within bounds
    if (row < n && col < n) {
        T_d[col * n + row] = A_d[row * n + col];
    }
}