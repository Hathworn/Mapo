#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized vecAdd kernel function
__global__ void vecAdd(unsigned int *A_d, unsigned int *B_d, unsigned int *C_d, int WORK_SIZE) {
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread within bounds
    if (id < WORK_SIZE) {
        C_d[id] = A_d[id] + B_d[id];
    }
}