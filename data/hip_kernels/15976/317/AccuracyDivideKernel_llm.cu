#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread block reduction to improve performance
    __shared__ float partialSum;
    if (threadIdx.x == 0) {
        partialSum = 0.0f;
    }
    __syncthreads();

    // Each thread computes a part of accuracy sum
    atomicAdd(&partialSum, accuracy[threadIdx.x]);

    __syncthreads();

    // Only one thread divides the result
    if (threadIdx.x == 0) {
        *accuracy = partialSum / N;
    }
}