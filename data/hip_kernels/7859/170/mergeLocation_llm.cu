#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    __shared__ short2 loc_shared[256];  // Declare shared memory for coalesced memory access
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tid = threadIdx.x;

    if (ptidx < npoints)
    {
        loc_shared[tid] = loc_[ptidx];  // Load data into shared memory
        __syncthreads();  // Synchronize to ensure all data is loaded

        x[ptidx] = loc_shared[tid].x * scale;  // Use shared memory for coalesced access
        y[ptidx] = loc_shared[tid].y * scale;
    }
}