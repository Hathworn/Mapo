#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Optimize boundary check by returning immediately if out of bounds.
    if (i >= h || j >= w) return;

    const int pos = i * s + j;

    float scale = normalization_factor[pos];

    // Optimize the conditional operation.
    float invScale = rsqrtf(scale * scale + 1e-8f);

    image[pos] *= invScale;
}