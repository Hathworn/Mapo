#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize empty kernel by removing unsued arguments or blocks
__global__ void sync_conv_groups() {
    // No operations needed as the kernel is empty
}