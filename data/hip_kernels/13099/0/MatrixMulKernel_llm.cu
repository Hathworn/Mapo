#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulKernel(float* d_M, float* d_N, float* d_P, int Width) 
{
    // Calculate the row index of the d_P element and d_M
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    // Calculate the column index of d_P and d_N
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for better memory access patterns
    __shared__ float s_M[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float s_N[BLOCK_SIZE][BLOCK_SIZE];

    float Pvalue = 0;

    for (int k = 0; k < (Width + BLOCK_SIZE - 1) / BLOCK_SIZE; ++k) {
        // Load data into shared memory
        if ((Row < Width) && (k * BLOCK_SIZE + threadIdx.x < Width))
            s_M[threadIdx.y][threadIdx.x] = d_M[Row * Width + k * BLOCK_SIZE + threadIdx.x];
        else
            s_M[threadIdx.y][threadIdx.x] = 0.0;

        if ((Col < Width) && (k * BLOCK_SIZE + threadIdx.y < Width))
            s_N[threadIdx.y][threadIdx.x] = d_N[(k * BLOCK_SIZE + threadIdx.y) * Width + Col];
        else
            s_N[threadIdx.y][threadIdx.x] = 0.0;

        __syncthreads();

        // Perform partial computation
        for (int n = 0; n < BLOCK_SIZE; ++n)
            Pvalue += s_M[threadIdx.y][n] * s_N[n][threadIdx.x];
        
        __syncthreads();
    }

    // Store the result in the global memory
    if ((Row < Width) && (Col < Width))
        d_P[Row * Width + Col] = Pvalue;
}