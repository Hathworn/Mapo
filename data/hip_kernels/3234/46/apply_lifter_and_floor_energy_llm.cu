#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {

    // Calculate thread and frame indices
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    float *feats = features + frame * ldf;

    // Apply lifter coefficients
    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += blockDim.x) { // Use blockDim.x instead of CU1DBLOCK for flexibility
            feats[c] *= lifter_coeffs[c]; // Directly multiply and assign for clarity
        }
    }

    // Apply energy flooring by the first thread of each block
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = log(energy_floor);

        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        feats[0] = energy;
    }
}