#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel( int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    float cur_weight = *cur_tot_weight;

    float new_num_frames = old_num_frames + cur_weight;
    float prior_scale_change = 1.0f;

    if (max_count != 0.0f) {
        float old_prior_scale = fmaxf(old_num_frames, (float)max_count) / max_count;
        float new_prior_scale = fmaxf(new_num_frames, (float)max_count) / max_count;
        prior_scale_change += new_prior_scale - old_prior_scale;
    }

    // Optimize loop by improving calculation with shared memory
    for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        atomicAdd(&quadratic[diag_idx], prior_scale_change);  // Use atomic operation to avoid race conditions
    }

    // Utilize potential unused threads for linear array update
    __shared__ float linear_shared;  // Shared variable to accumulate updates
    if (threadIdx.x == 0) {
        linear_shared = 0.0f;
    }
    __syncthreads();
    
    // Each block will only call this once
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        atomicAdd(&linear_shared, prior_offset * prior_scale_change);
    }
    __syncthreads();

    if (threadIdx.x == 0 && blockIdx.x == 0) {
        atomicAdd(&linear[0], linear_shared);
    }
}