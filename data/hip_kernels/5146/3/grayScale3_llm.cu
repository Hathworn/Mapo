#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void grayScale3(uchar3 *input, uchar3 *output, int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Check within image bounds
    if (x < width && y < height) { 
        int tid = y * width + x;

        // Calculate grayscale value using integer arithmetic for performance
        unsigned char grayValue = (input[tid].x + input[tid].y + input[tid].z) / 3;

        // Assign grayscale value to all color channels
        output[tid] = make_uchar3(grayValue, grayValue, grayValue);
    }
}