#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hsv2rgb(float *inputH, float *inputS, float *inputV, uchar3 *output, int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int tid = y * width + x;

    // Ensure within image bounds
    if (x < width && y < height) {
        float H = inputH[tid];
        float S = inputS[tid];
        float V = inputV[tid];

        // Precompute common values
        float d = H / 60.0f;
        int hi = (int)d % 6;
        float f = d - hi;
        float l = V * (1.0f - S);
        float m = V * (1.0f - f * S);
        float n = V * (1.0f - (1.0f - f) * S);

        // Use switch to reduce repeated comparisons
        switch(hi) {
            case 0:
                output[tid] = make_uchar3((int)(V * 255), (int)(n * 255), (int)(l * 255));
                break;
            case 1:
                output[tid] = make_uchar3((int)(m * 255), (int)(V * 255), (int)(l * 255));
                break;
            case 2:
                output[tid] = make_uchar3((int)(l * 255), (int)(V * 255), (int)(n * 255));
                break;
            case 3:
                output[tid] = make_uchar3((int)(l * 255), (int)(m * 255), (int)(V * 255));
                break;
            case 4:
                output[tid] = make_uchar3((int)(n * 255), (int)(l * 255), (int)(V * 255));
                break;
            case 5:
                output[tid] = make_uchar3((int)(V * 255), (int)(l * 255), (int)(m * 255));
                break;
        }
    }
}