#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void blur(uchar3 *input, uchar3 *output, int width, int height) {
    __shared__ int matrix[7][7];
    if (threadIdx.x < 7 && threadIdx.y < 7) {
        const int mat_init[7][7] = {
            {0, 0, 1, 2, 1, 0, 0},
            {0, 3, 13, 22, 13, 3, 0},
            {1, 3, 59, 97, 59, 13, 1},
            {2, 22, 97, 159, 97, 22, 2},
            {1, 3, 59, 97, 59, 3, 1},
            {0, 3, 13, 22, 13, 3, 0},
            {0, 0, 1, 2, 1, 0, 0}
        };
        matrix[threadIdx.x][threadIdx.y] = mat_init[threadIdx.x][threadIdx.y];
    }
    __syncthreads();
    
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    int tid = y * width + x;
    
    if (x >= 3 && x < width-3 && y >= 3 && y < height-3) {
        int outputTemp = 0;
        int sommeCoef = 0;
        
        for (int i = 0; i < 7; i++) {
            for (int j = 0; j < 7; j++) {
                // Streamline calculations by using register storage for input pixel
                uchar3 currPixel = input[(y-3+i)*width + (x-3+j)];
                outputTemp += currPixel.x * matrix[j][i];
                sommeCoef += matrix[j][i];
            }
        }
        // Optimize coefficient division
        output[tid].x = __float2int_rn(outputTemp / static_cast<float>(sommeCoef));
        output[tid].z = output[tid].y = output[tid].x;
    }
}