#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void grayScale2(uchar3 *input, uchar3 *output, int width, int height) {

    // Calculate pixel indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if pixel is within bounds
    if (x < width && y < height) {
        int tid = y * width + x;

        // Compute grayscale value
        unsigned char gray = (input[tid].x + input[tid].y + input[tid].z) / 3;

        // Assign grayscale value
        output[tid].x = gray;
        output[tid].y = gray;
        output[tid].z = gray;
    }
}