#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blendingGray(uchar3 *input, uchar3 *input2, uchar3 *output, int width, int height, float coefficient) {
    // Calculate thread's row and column
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within image bounds
    if (x < width && y < height) {
        // Compute unique thread index
        int tid = y * width + x;

        // Calculate threshold index
        int prodfin = (int)(coefficient * (float)(width * height));

        // Conditional output assignment
        uchar3& targetPix = (tid <= prodfin) ? input[tid] : input2[tid];
        output[tid].x = targetPix.x;
        output[tid].y = output[tid].z = output[tid].x;
    }
}