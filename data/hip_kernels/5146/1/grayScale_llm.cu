#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void grayScale(uchar3 *input, uchar3 *output) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned char gray = (input[tid].x + input[tid].y + input[tid].z) / 3; // Pre-calculate grayscale value
    output[tid] = make_uchar3(gray, gray, gray); // Direct assignment using make_uchar3
}