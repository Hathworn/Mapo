#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAddInt32(int32_t* in0, int32_t* in1, int32_t* out, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int step = blockDim.x * gridDim.x;  // Calculate step size for loop unrolling
    for(int i = tid; i < cnt; i += step) {  // Loop unrolling optimization
        out[i] = in0[i] + in1[i];
    }
}