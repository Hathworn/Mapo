#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecSubFp32(float* __restrict__ in0, float* __restrict__ in1, float* __restrict__ out, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < cnt) {
        out[tid] = __fsub_rn(in0[tid], in1[tid]); // Use fast sub intrinsic
    }
}