#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecSubInt32(int32_t* __restrict__ in0, int32_t* __restrict__ in1, int32_t* __restrict__ out, int cnt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop for better efficiency
    for (int i = tid; i < cnt; i += blockDim.x * gridDim.x) {
        out[i] = in0[i] - in1[i];
    }
}