#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAddFp32(float* __restrict__ in0, float* __restrict__ in1, float* __restrict__ out, int cnt)
{
    // Use local variable to minimize global memory access latency
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < cnt) {
        float a = in0[tid];
        float b = in1[tid];
        out[tid] = a + b;
    }
}