#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop unrolling for more efficient memory access
    int stride = blockDim.x * gridDim.x;
    while (idx < num_elements) {
        int base_idx = idx * 3;
        to_normalize[base_idx] = batch_index;
        to_normalize[base_idx + 1] = class_index;
        to_normalize[base_idx + 2] = static_cast<int64_t>(original[idx]);
        idx += stride;
    }
}