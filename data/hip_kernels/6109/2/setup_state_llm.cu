#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Using blockIdx.x to allow multiple states initialization in parallel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}