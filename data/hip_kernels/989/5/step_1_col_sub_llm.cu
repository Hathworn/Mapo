#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define klog2(n) ((n<8)?2:((n<16)?3:((n<32)?4:((n<64)?5:((n<128)?6:((n<256)?7:((n<512)?8:((n<1024)?9:((n<2048)?10:((n<4096)?11:((n<8192)?12:((n<16384)?13:0))))))))))))

#ifndef DYNAMIC
#define MANAGED __managed__
#define dh_checkCuda checkCuda
#define dh_get_globaltime get_globaltime
#define dh_get_timer_period get_timer_period
#else
#define dh_checkCuda d_checkCuda
#define dh_get_globaltime d_get_globaltime
#define dh_get_timer_period d_get_timer_period
#define MANAGED
#endif

#define kmin(x,y) ((x<y)?x:y)
#define kmax(x,y) ((x>y)?x:y)

#ifndef USE_TEST_MATRIX
#ifdef _n_
const int n = _n_;
const int range = _range_;
const int user_n = n;
const int n_tests = 100;
#else
const int user_n = 1000;
const int n = 1<<(klog2(user_n)+1);
const int range = n;
const int n_tests = 10;
#endif

const int log2_n = klog2(n);
const int n_threads = kmin(n,64);
const int n_threads_reduction = kmin(n, 256);
const int n_blocks_reduction = kmin(n, 256);
const int n_threads_full = kmin(n, 512);
const int seed = 45345;

#else
const int n = 4;
const int log2_n = 2;
const int n_threads = 2;
const int n_threads_reduction = 2;
const int n_blocks_reduction = 2;
const int n_threads_full = 2;
#endif

const int n_blocks = n / n_threads;
const int n_blocks_full = n * n / n_threads_full;
const int row_mask = (1 << log2_n) - 1;
const int nrows = n, ncols = n;
const int max_threads_per_block = 1024;
const int columns_per_block_step_4 = 512;
const int n_blocks_step_4 = kmax(n / columns_per_block_step_4, 1);
const int data_block_size = columns_per_block_step_4 * n;
const int log2_data_block_size = log2_n + klog2(columns_per_block_step_4);

#ifndef CHAR_DATA_TYPE
typedef int data;
#define MAX_DATA INT_MAX
#define MIN_DATA INT_MIN
#else
typedef unsigned char data;
#define MAX_DATA 255
#define MIN_DATA 0
#endif

#ifndef USE_TEST_MATRIX
data h_cost[ncols][nrows];
#else
data h_cost[n][n] = { { 1, 2, 3, 4 }, { 2, 4, 6, 8 }, { 3, 6, 9, 12 }, { 4, 8, 12, 16 } };
#endif
int h_column_of_star_at_row[nrows];
int h_zeros_vector_size;
int h_n_matches;
bool h_found;
bool h_goto_5;

__device__ data slack[nrows*ncols];
__device__ data min_in_rows[nrows];
__device__ data min_in_cols[ncols];
__device__ int zeros[nrows*ncols];
__device__ int zeros_size_b[n_blocks_step_4];

__device__ int row_of_star_at_column[ncols];
__device__ int column_of_star_at_row[nrows];
__device__ int cover_row[nrows];
__device__ int cover_column[ncols];
__device__ int column_of_prime_at_row[nrows];
__device__ int row_of_green_at_column[ncols];

__device__ data max_in_mat_row[nrows];
__device__ data min_in_mat_col[ncols];
__device__ data d_min_in_mat_vect[n_blocks_reduction];
__device__ data d_min_in_mat;

MANAGED __device__ int zeros_size;
MANAGED __device__ int n_matches;
MANAGED __device__ bool goto_5;
MANAGED __device__ bool repeat_kernel;
#if defined(DEBUG) || defined(_DEBUG)
MANAGED __device__ int n_covered_rows;
MANAGED __device__ int n_covered_columns;
#endif

__shared__ extern data sdata[];

__global__ void step_1_col_sub()
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int c = i >> log2_n;
    
    if (i < nrows * ncols) {
        // Avoid accessing out of bounds
        slack[i] -= min_in_cols[c]; // Efficient column subtraction
    }
    
    if (i == 0) zeros_size = 0;
    if (i < n_blocks_step_4) zeros_size_b[i] = 0;
}