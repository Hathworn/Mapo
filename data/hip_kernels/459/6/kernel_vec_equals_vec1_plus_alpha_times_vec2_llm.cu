#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void kernel_vec_equals_vec1_plus_alpha_times_vec2(double *vec, double *vec1, double alpha, double *d_a1, double *vec2, int numElements)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Single if condition to ensure valid gid
    if (gid < numElements) {
        double a = alpha * (d_a1 ? *d_a1 : 1.0); // Inline check for d_a1 and calculate a

        vec[gid] = vec1[gid] + a * vec2[gid];    // Perform computation
    }
}