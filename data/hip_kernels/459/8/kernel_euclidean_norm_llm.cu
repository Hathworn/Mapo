#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Launch this kernel with a single block and numElements threads
__global__ void kernel_euclidean_norm(const double *vec, int numElements, double *answer)
{
    extern __shared__ double square[];

    int i = threadIdx.x;

    // Compute square of each element and store
    if (i < numElements) {
        square[i] = vec[i] * vec[i];
    } else {
        square[i] = 0.0; // Ensure no garbage values for unused threads
    }

    __syncthreads();

    // Parallel reduction to sum squares
    for (int stride = 1; stride < numElements; stride *= 2) {
        int index = 2 * stride * i;

        if (index < numElements) {
            square[index] += square[index + stride];
        }

        __syncthreads();
    }

    // Thread 0 calculates and writes the final result
    if (i == 0) {
        *answer = sqrt(square[0]);
    }
}