#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_vec_equals_minus_vec1(double *vec, double *vec1, int numElements)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure gid is within bounds
    if (gid < numElements) {
        vec[gid] = -vec1[gid];
    }
}