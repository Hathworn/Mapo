#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void doubleArray2floatArray(const double * doubleArray, float* floatArray, const int size) {
    // Calculate the global thread index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use CUDA warp shuffle to optimize memory access
    if (i < size) {
        // Perform type casting in a more efficient way
        floatArray[i] = __double2float_rd(doubleArray[i]);
    }
}