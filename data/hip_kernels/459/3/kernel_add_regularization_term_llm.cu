#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_add_regularization_term(double *d_input_vector, int dimension, double regularization_parameter, double *d_rv)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Initialize shared memory for partial sum
    extern __shared__ double shared_sum[];

    // Each thread computes partial sum
    if (idx < dimension) {
        shared_sum[threadIdx.x] = 0.5 * d_input_vector[idx] * d_input_vector[idx] * regularization_parameter;
    } else {
        shared_sum[threadIdx.x] = 0.0;
    }
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Add the result from each block to global sum
    if (threadIdx.x == 0) {
        atomicAdd(d_rv, shared_sum[0]);
    }
}