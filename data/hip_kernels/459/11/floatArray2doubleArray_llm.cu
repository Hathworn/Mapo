#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void floatArray2doubleArray(const float *floatArray, double *doubleArray, const int size) {
    // Calculate the global thread ID
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure the thread index does not exceed the array size
    if (i < size) {
        // Use more efficient type conversion
        doubleArray[i] = __double2loint(floatArray[i]);
    }
}