#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_mult_vector_by_number(double *vec, double alpha, int numElements)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by using grid-stride loops to handle large data efficiently
    for (int i = gid; i < numElements; i += blockDim.x * gridDim.x) {
        vec[i] *= alpha;
    }
}