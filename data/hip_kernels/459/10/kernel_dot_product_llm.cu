#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_dot_product(const double *vec1, const double *vec2, int numElements, double *answer)
{
    extern __shared__ double products[];

    int i = threadIdx.x;

    // Load product into shared memory
    products[i] = vec1[i] * vec2[i];
    __syncthreads();

    // Reduce using a parallel reduction
    for (int stride = 1; stride < numElements; stride *= 2) {
        if (i % (2 * stride) == 0 && i + stride < numElements) {
            products[i] += products[i + stride];
        }
        __syncthreads();
    }

    // Store the result
    if (i == 0) {
        *answer = products[0];
    }
}