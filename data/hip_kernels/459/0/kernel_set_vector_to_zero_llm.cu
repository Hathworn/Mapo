#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_set_vector_to_zero(double *d_vec, int dimension)
{
    int gid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID

    if (gid < dimension) {
        d_vec[gid] = 0.0; // Use 0.0 for consistency with double precision
    }
}