#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
    // Calculate the global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a strided loop for better performance on larger arrays
    for (int i = idx; i < size; i += blockDim.x * gridDim.x) {
        r[i] = x[i] * y[i] + z[i];
    }
}