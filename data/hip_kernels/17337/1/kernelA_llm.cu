#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernelA(float* r, float* x, float* y, float* z, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < size) { // Ensure index is within bounds
        r[i] = x[i] * y[i] + z[i];
    }
}