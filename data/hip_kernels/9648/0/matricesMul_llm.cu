#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define rows 1000
#define cols 1000

// CUDA kernel. Each thread takes care of one element of c

__global__ void matricesMul(double *m1, double *m2, double *m3)
{
    // Use shared memory to optimize memory access
    __shared__ double shared_m1[32][32];
    __shared__ double shared_m2[32][32];

    // Calculate row and column index
    int ti = blockIdx.y * blockDim.y + threadIdx.y;
    int tj = blockIdx.x * blockDim.x + threadIdx.x;

    double data = 0.0;
    
    // Loop over sub-matrices
    for (int sub = 0; sub < (rows + 31) / 32; ++sub) {
        if (ti < rows && (sub * 32 + threadIdx.x) < cols) {
            shared_m1[threadIdx.y][threadIdx.x] = m1[ti * rows + sub * 32 + threadIdx.x];
        } else {
            shared_m1[threadIdx.y][threadIdx.x] = 0.0;
        }
        
        if ((sub * 32 + threadIdx.y) < rows && tj < cols) {
            shared_m2[threadIdx.y][threadIdx.x] = m2[(sub * 32 + threadIdx.y) * cols + tj];
        } else {
            shared_m2[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads();

        // Multiply and accumulate
        for (int k = 0; k < 32; ++k) {
            data += shared_m1[threadIdx.y][k] * shared_m2[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Ensure we do not go out of bounds before writing the result
    if (ti < rows && tj < cols) {
        m3[ti * rows + tj] = data;
    }
}