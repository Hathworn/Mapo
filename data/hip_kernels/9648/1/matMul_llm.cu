#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matMul(unsigned char *image, unsigned char *resImage, int rows, int cols) {
    // Calculate unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize boundary check and loop unrolling
    int totalPixels = rows * cols;
    if (idx < totalPixels) {
        int pos = idx * chanDepth;
        unsigned char blue = image[pos + BLUE];
        unsigned char green = image[pos + GREEN];
        unsigned char red = image[pos + RED];

        // Apply one transformation and assign 
        resImage[pos + BLUE] = blue * 2;
        resImage[pos + GREEN] = green * 2;
        resImage[pos + RED] = red * 2;
    }
}