#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Use a single loop for reduction and compute partial inner product.
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Reduction using warp shuffle to avoid multiple __syncthreads
    if (idx < 32) {
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            tmp += __shfl_down_sync(0xFFFFFFFF, tmp, offset);
        }
        if (idx == 0) {
            reduction_buffer[0] = tmp;
        }
    }
    __syncthreads();

    // Compute gradient using reduced value.
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}