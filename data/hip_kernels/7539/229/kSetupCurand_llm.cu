#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;

    // Use a unique seed for each block combined with thread index to increase randomness
    unsigned long long blockSeed = seed + blockIdx.x;
    hiprand_init(blockSeed, tidx, 0, &state[tidx]);
}