#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    
    // Calculate total elements once to avoid repetitive calculation
    const uint numEls = tgtWidth * tgtHeight;
    
    // Use loop unrolling for better performance if possible
    for (uint i = idx; i < numEls; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Load once, store once
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}