#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Optimize the computation by using a ternary operator and unrolling loop.
        correctProbs[tx] = (labelp != maxp) ? 0.0f : 1.0f / float([&](){
            int numMax = 0;
            for (int i = 0; i < numOut; i += 4) { // Unroll loop for efficiency.
                numMax += (probs[i * numCases + tx] == maxp);
                if (i + 1 < numOut) numMax += (probs[(i + 1) * numCases + tx] == maxp);
                if (i + 2 < numOut) numMax += (probs[(i + 2) * numCases + tx] == maxp);
                if (i + 3 < numOut) numMax += (probs[(i + 3) * numCases + tx] == maxp);
            }
            return numMax;
        }());
    }
}