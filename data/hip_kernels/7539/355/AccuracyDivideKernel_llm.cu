#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread ID check to ensure only one thread performs the division
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}