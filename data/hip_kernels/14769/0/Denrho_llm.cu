#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int BLOCK_SIZE_X = 26;
const int BLOCK_SIZE_Y = 26;
const float w1 = 4.0/9.0, w2 = 1.0/9.0, w3 = 1.0/36.0;
const float Amp2 = 0.1, Width = 10, omega = 1;

__global__ void Denrho(float* u_d, float* f_d, int ArraySizeX, int ArraySizeY)
{
    // Use shared memory to reduce global memory access
    __shared__ float f_shared[BLOCK_SIZE_X][BLOCK_SIZE_Y][9];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x*(BLOCK_SIZE_X-2);
    int by = blockIdx.y*(BLOCK_SIZE_Y-2);
    int x = tx + bx;
    int y = ty + by;

    // Load data into shared memory
    if (x < ArraySizeX && y < ArraySizeY) {
        for (int i = 0; i < 9; i++) {
            f_shared[tx][ty][i] = f_d[x*ArraySizeY*9 + y*9 + i];
        }
    }
    __syncthreads();

    // Aggregate the results using shared memory
    if (x < ArraySizeX && y < ArraySizeY) {
        float sum = 0;
        for (int i = 0; i < 9; i++) {
            sum += f_shared[tx][ty][i];
        }
        u_d[x*ArraySizeY + y] = sum;
    }
}