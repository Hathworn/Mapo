#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int BLOCK_SIZE_X = 26;
const int BLOCK_SIZE_Y = 26;
const float w1 = 4.0/9.0, w2 = 1.0/9.0, w3 = 1.0/36.0;
const float Amp2 = 0.1, Width = 10, omega = 1;

__global__ void iteration(float* f_d, int ArraySizeX, int ArraySizeY)
{
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x*(BLOCK_SIZE_X-2);
    int by = blockIdx.y*(BLOCK_SIZE_Y-2);
    int x = tx + bx;
    int y = ty + by;
    float n, ux, uy, uxx, uyy, usq, Fx, Fy, Fsq;
    __shared__ float f_sh[BLOCK_SIZE_X][BLOCK_SIZE_Y][9];

    // Load data into shared memory
    #pragma unroll
    for (int i = 0; i < 9; i++)
        f_sh[tx][ty][i] = f_d[x*ArraySizeY*9 + y*9 + i];

    __syncthreads();

    // Compute macroscopic quantities
    n = 0.0f;
    ux = 0.0f;
    uy = 0.0f;
    #pragma unroll
    for (int i = 0; i < 9; i++) {
        n += f_sh[tx][ty][i];
    }
    ux = (f_sh[tx][ty][1] - f_sh[tx][ty][2] + f_sh[tx][ty][5] - f_sh[tx][ty][6] - f_sh[tx][ty][7] + f_sh[tx][ty][8]) / n;
    uy = (f_sh[tx][ty][3] - f_sh[tx][ty][4] + f_sh[tx][ty][5] + f_sh[tx][ty][6] - f_sh[tx][ty][7] - f_sh[tx][ty][8]) / n;
    uxx = ux * ux;
    uyy = uy * uy;
    usq = uxx + uyy;

    // Forcing terms
    Fx = 0;
    Fy = 0;
    Fsq = 0.0f;

    // Perform collision step
    float feq, forceTerm, *f;
    f = f_sh[tx][ty];
    feq = w1 * n * (1.0f - 1.5f * usq);
    funroll
    for (int i = 0; i < 9; i++) {
        forceTerm = (i < 5) ? w2 * (3.0f * Fx * ux + 3.0f * Fy * uy) : w3 * (3.0f * Fx * ux + 3.0f * Fy * uy);
        f[i] += omega * (feq - f[i]) + forceTerm;
    }
    __syncthreads();

    // Perform stream step
    if (tx > 0 && tx < BLOCK_SIZE_X - 1 && ty > 0 && ty < BLOCK_SIZE_Y - 1) {
        f_d[x*ArraySizeY*9 + y*9]   = f[0];
        f_d[x*ArraySizeY*9 + y*9+2] = f_sh[tx+1][ty][2];
        f_d[x*ArraySizeY*9 + y*9+1] = f_sh[tx-1][ty][1];
        f_d[x*ArraySizeY*9 + y*9+4] = f_sh[tx][ty+1][4];
        f_d[x*ArraySizeY*9 + y*9+3] = f_sh[tx][ty-1][3];
        f_d[x*ArraySizeY*9 + y*9+7] = f_sh[tx+1][ty+1][7];
        f_d[x*ArraySizeY*9 + y*9+5] = f_sh[tx-1][ty-1][5];
        f_d[x*ArraySizeY*9 + y*9+6] = f_sh[tx+1][ty-1][6];
        f_d[x*ArraySizeY*9 + y*9+8] = f_sh[tx-1][ty+1][8];
    }
    __syncthreads();

    // Apply periodic boundary conditions
    if (x == 0)
        for (int i = 0; i < 9; i++)
            f_d[x*ArraySizeY*9 + y*9 + i] = f_d[(ArraySizeX-2)*ArraySizeY*9 + y*9 + i];
    if (x == ArraySizeX-1)
        for (int i = 0; i < 9; i++)
            f_d[x*ArraySizeY*9 + y*9 + i] = f_d[ArraySizeY*9 + y*9 + i];
    if (y == 0)
        for (int i = 0; i < 9; i++)
            f_d[x*ArraySizeY*9 + y*9 + i] = f_d[x*ArraySizeY*9 + (ArraySizeY-2)* 9 + i];
    if (y == ArraySizeY-1)
        for (int i = 0; i < 9; i++)
            f_d[x*ArraySizeY*9 + y*9 + i] = f_d[x*ArraySizeY*9 + 9 + i];
}