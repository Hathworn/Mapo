#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void detect_edges(unsigned char* in, unsigned char* out, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x; // Calculate x
    int y = blockIdx.y * blockDim.y + threadIdx.y; // Calculate y

    if (x >= width || y >= height) return; // Check bounds

    int i = y * width + x; // Flatten y and x to get 1D index
    if (x == 0 || y == 0 || x == width - 1 || y == height - 1) {
        out[i] = 0; // Handle border pixels
    } else {
        int b = i + width;
        int d = i - 1;
        int f = i + 1;
        int h = i - width;

        int r = (in[i] * 4) + (in[b] * -1) + (in[d] * -1) + (in[f] * -1) + (in[h] * -1); // Convolution

        out[i] = (r > 0) ? 255 : 0; // Set output
    }
}