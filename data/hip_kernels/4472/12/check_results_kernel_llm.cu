#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void check_results_kernel(int n, double correctvalue, double *x)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        double xi = x[i]; // Load x[i] once
        if (xi != correctvalue) // Compare with preloaded value
        {
            printf("ERROR at index = %d, expected = %f, actual: %f\n", i, correctvalue, xi);
        }
    }
}