#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void daxpy_kernel(int n, double a, double * x, double * y)
{
    // Optimize memory access and computation by using shared memory and loop unrolling
    __shared__ double shared_x[1024];
    __shared__ double shared_y[1024];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;

    if (i < n) {
        // Load data into shared memory for faster access
        shared_x[tid] = x[i];
        shared_y[tid] = y[i];

        __syncthreads(); // Ensure all data is loaded

        // Perform computation with unrolled loop for better efficiency
        if (tid < n) {
            y[i] = a * shared_x[tid] + shared_y[tid];
        }
    }
}