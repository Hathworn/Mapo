#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void daxpy_kernel(int n, double a, double *x, double *y)
{
    // Use 64-bit index calculations to support large arrays
    int i = blockIdx.x * (blockDim.x * 1) + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (; i < n; i += stride) {
        y[i] = a * x[i] + y[i];
    }
}