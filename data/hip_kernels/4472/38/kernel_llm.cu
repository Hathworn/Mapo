#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float *a, int offset)
{
    int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
    float x = static_cast<float>(i);

    // Use sincosf to compute sin and cos simultaneously for better performance
    float s, c;
    sincosf(x, &s, &c);
    
    // Use fmaf for fused multiply-add operation to improve accuracy and performance
    a[i] = fmaf(s, s, c * c);
}