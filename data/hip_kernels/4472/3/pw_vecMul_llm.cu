#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_vecMul(float *y, float *a, float *b, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Use a stride loop to process more elements per thread, improving load balance
    for (int i = idx; i < n; i += stride) {
        y[i] = a[i] * b[i];
    }
}