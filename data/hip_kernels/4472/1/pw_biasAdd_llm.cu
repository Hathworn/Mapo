#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_biasAdd(float *y, float *bias, int n, int nBias) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        // Cache the bias value to reduce global memory access.
        float localBias = bias[i % nBias];
        y[i] += localBias;
    }
}