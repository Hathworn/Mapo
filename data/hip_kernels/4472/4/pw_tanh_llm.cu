#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_tanh(float *y, float *a, int n) {
    // Use shared memory for faster access
    extern __shared__ float shared_a[];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Load data into shared memory
    if (i < n) {
        shared_a[tid] = a[i];
        __syncthreads();

        // Compute tanh using shared memory
        y[i] = tanh(shared_a[tid]);
    }
}