#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copySharedMem(float *odata, const float *idata)
{
    __shared__ float tile[TILE_DIM * TILE_DIM];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    // Load data into shared memory with coalesced accesses
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x] = idata[(y+j)*width + x];

    __syncthreads();

    // Write data from shared memory to output with coalesced accesses
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y+j)*width + x] = tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x];
}