#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy(float *odata, const float *idata)
{
    // Use shared memory to optimize memory access
    __shared__ float tile[TILE_DIM][TILE_DIM];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    // Load data into shared memory
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        tile[threadIdx.y + j][threadIdx.x] = idata[(y + j) * width + x];
    }
    __syncthreads(); // Ensure all data is loaded

    // Store data from shared memory to global memory
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        odata[(y + j) * width + x] = tile[threadIdx.y + j][threadIdx.x];
    }
}