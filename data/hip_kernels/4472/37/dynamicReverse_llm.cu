#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dynamicReverse(int *d, int n)
{
    extern __shared__ int s[];
    int t = threadIdx.x;
    int b = blockIdx.x * blockDim.x;
    
    // Calculate the index for the reversed array
    int tr = n - b - t - 1;
    // Load data from global to shared memory
    s[t] = d[b + t];
    __syncthreads();
    // Store reversed data back to global memory
    d[b + t] = s[tr - b];
}