#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_vecAdd(float *y, float *a, float *b, int n) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use inline computation to prevent excess memory access
    if (i < n) y[i] = __fadd_rd(a[i], b[i]);  // Fast addition for improved performance
}