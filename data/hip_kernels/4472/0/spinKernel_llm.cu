#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void spinKernel(unsigned long long timeout_clocks = 100000ULL)
{
    // Use a more efficient calculation for the loop condition
    unsigned long long start_time = clock64();

    // Simplified polling for timeout
    while (clock64() - start_time <= timeout_clocks) {
        // Empty loop body for spin wait
    }
}