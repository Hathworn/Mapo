#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_data_kernel(int n, double* x)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Using a stride loop improves memory access patterns
    for (; i < n; i += blockDim.x * gridDim.x)
    {
        x[i] = n - i;
    }
}