#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNaive(float *odata, const float *idata)
{
    // Use shared memory to optimize data access
    __shared__ float tile[TILE_DIM][TILE_DIM];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    // Load data into shared memory
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        tile[threadIdx.y + j][threadIdx.x] = idata[(y + j) * width + x];
    }
    __syncthreads();

    // Write data from shared memory to output
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
        odata[(y + j) * width + x] = tile[threadIdx.x][threadIdx.y + j];
    }
}