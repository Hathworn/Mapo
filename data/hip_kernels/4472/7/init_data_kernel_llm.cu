#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_data_kernel(int n, double* x) {
    // Use static shared memory and loop unrolling for better memory access
    int total_threads = gridDim.x * blockDim.x;
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += total_threads) {
        x[i] = n - i;
    }
}