#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void check_results_kernel(int n, double correctvalue, double *x)
{
    // Calculate global thread index in grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a conditional statement to limit unnecessary operations
    if (i < n && x[i] != correctvalue)
    {
        printf("ERROR at index = %d, expected = %f, actual: %f\n", i, correctvalue, x[i]);
    }
}