#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNoBankConflicts(float *odata, const float *idata)
{
    __shared__ float tile[TILE_DIM][TILE_DIM+1];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    // Load data into shared memory with coalesced access
    #pragma unroll
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        tile[threadIdx.y + j][threadIdx.x] = idata[(y + j) * width + x];

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    // Store transposed data into global memory
    #pragma unroll
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
        odata[(y + j) * width + x] = tile[threadIdx.x][threadIdx.y + j];
}