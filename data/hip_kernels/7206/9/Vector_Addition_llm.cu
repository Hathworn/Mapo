#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Vector_Addition(const int *dev_a, const int *dev_b, int *dev_c)
{
    // Get the id of thread within the grid
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < N) // Check the boundary condition for the threads
        dev_c[tid] = dev_a[tid] + dev_b[tid];
}