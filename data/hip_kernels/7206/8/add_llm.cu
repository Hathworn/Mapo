#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Use blockIdx and blockDim to calculate unique thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use thread index to access different elements in array
    c[index] = a[index] + b[index];
}