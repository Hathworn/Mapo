#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cu_init(unsigned long long seed, hiprandState_t * states_d, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= size) return; // Early exit if thread index is out of bounds
    hiprand_init(seed, idx, 0, &states_d[idx]);
}