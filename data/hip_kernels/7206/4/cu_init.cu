#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cu_init(unsigned long long seed, hiprandState_t * states_d, size_t size) {
int idx = threadIdx.x + blockIdx.x * blockDim.x;
if(idx < size) {
hiprand_init(seed, idx, 0, &states_d[idx]);
}
}