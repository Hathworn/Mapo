#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Vector_Addition(const int *dev_a, const int *dev_b, int *dev_c)
{
    // Get the global thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by checking boundary condition once and using for loop
    for (; tid < N; tid += blockDim.x * gridDim.x)
    {
        dev_c[tid] = dev_a[tid] + dev_b[tid];
    }
}