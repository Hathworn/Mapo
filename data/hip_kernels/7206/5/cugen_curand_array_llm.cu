#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cugen_curand_array(hiprandState_t *states_d, int *array_d, size_t size) {
    // Calculate the thread's global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure that the index is within bounds
    if (idx < size) {
        // Use __ldg to load state for potential cache performance improvement
        hiprandState_t state = __ldg(&states_d[idx]); 
        
        // Generate a random number using the state
        int r = hiprand_uniform(&state) * 100;
        
        // Store the result in the output array
        array_d[idx] = r;
        
        // Update the state back to global memory
        states_d[idx] = state;
    }
}