#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Vector_Addition(const int *dev_a, const int *dev_b, int *dev_c)
{
    // Calculate global thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Access global memory using global thread ID, check boundary condition
    if (tid < N)
        dev_c[tid] = dev_a[tid] + dev_b[tid];
}