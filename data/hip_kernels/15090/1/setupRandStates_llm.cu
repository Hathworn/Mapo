#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 100

__global__ void setupRandStates(hiprandState_t* state, unsigned int seed) {
    // Calculate global thread ID using built-in variables
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    // Each thread gets unique seed offset by thread ID
    hiprand_init(seed, thread_id, 0, &state[thread_id]);
}