#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"






#define N 100

__global__ void setupRandStates(hiprandState_t* state, unsigned int seed) {
unsigned block_id = blockIdx.y * gridDim.x + blockIdx.x;
int thread_id = threadIdx.x + block_id * blockDim.x;
// Each thread gets same seed, a different sequence number, no offset
hiprand_init(seed, thread_id, 0, &state[thread_id]);

}