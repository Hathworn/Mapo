#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];

    // Load input elements into shared memory
    tmp[threadIdx.x] = input[threadIdx.x + blockIdx.x * blockDim.x];
    __syncthreads();

    // Perform reduction using loop to minimize code repetition and syncthreads calls
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            tmp[threadIdx.x] += tmp[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (threadIdx.x == 0) {
        output[blockIdx.x] = tmp[0];
    }
}