#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    // Cache input vector in shared memory for faster access
    __shared__ double shared_vector_in[blockDim.x];
    
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    double out = 0.0;

    for (long j = threadIdx.x; j < dim_mn; j += blockDim.x) {
        shared_vector_in[threadIdx.x] = vector_in[j];
        __syncthreads(); // Synchronize to make sure all threads have loaded the data

        if (i < dim_mn) {
            for (long k = 0; k < blockDim.x && (j + k) < dim_mn; ++k) {
                out += matrix[i * dim_mn + j + k] * shared_vector_in[k];
            }
        }
        __syncthreads(); // Synchronize before next iteration
    }

    if (i < dim_mn) {
        vector_out[i] = out;
    }
}