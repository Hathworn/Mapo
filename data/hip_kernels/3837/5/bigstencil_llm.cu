#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bigstencil(int* in, int* out) {
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (i < N) {
        // Add 2 to input and store in output
        out[i] = in[i] + 2;
    }
}