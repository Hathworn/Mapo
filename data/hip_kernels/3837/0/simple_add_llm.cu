#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_add(float* __restrict__ a, const float* __restrict__ b, int n)
{
    // Calculate index using standard formula
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use conditional operator to limit out-of-bound access
    if(i < n) {
        a[i] += b[i]; // Simplified in-place addition
    }
}