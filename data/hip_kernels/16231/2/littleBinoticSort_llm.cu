#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Swap function
__device__ void swap(int &a, int &b){
    int t = a;
    a = b;
    b = t;
}

__global__ void littleBinoticSort(int* arr, int num, int numMax){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Set out-of-bound threads' data to INT_MAX
    if(tid >= num) arr[tid] = INT_MAX;

    __syncthreads();

    // Loop optimizations for sorting
    for(unsigned int i = 2; i <= numMax; i <<= 1){
        for(unsigned int j = i >> 1; j > 0; j >>= 1){
            unsigned int swapIdx = tid ^ j;

            // Ensure valid swap index and apply the sorting conditionally
            if(swapIdx < num && swapIdx > tid){
                bool tidIsEven = (tid & i) == 0;
                if((tidIsEven && arr[tid] > arr[swapIdx]) || (!tidIsEven && arr[tid] < arr[swapIdx])){
                    swap(arr[tid], arr[swapIdx]);
                }
            }

            __syncthreads();
        }
    }
}