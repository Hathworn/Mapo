#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;

    // Use shared memory to reduce global memory accesses for better performance
    __shared__ float2 shared_A[512];
    if (thread_id < half_length) {
        shared_A[thread_id] = reinterpret_cast<float2 *>(Ar)[thread_id];
    }
    __syncthreads();

    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        // ignore special case
        if (idx == 0) continue;

        float2 val = shared_A[idx];
        float ret = val.x * val.x + val.y * val.y;
        if (use_power) {
            Aw[idx] = ret;
        } else {
            Aw[idx] = sqrtf(ret);
        }
    }

    // handle special case with optimized calculations
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabs(real);
            Aw[half_length] = fabs(im);
        }
    }
}