#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // Calculate global row index
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global column index

    if (row < num_rows && col < num_cols) {  // Ensure indexes are within bounds
        float f = feats[row * ldf + col];  // Optimized memory access
        feats_sq[row * lds + col] = f * f;  // Compute square and store
    }
}