#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    const float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    
    // Optimize loop by unrolling and reducing branch divergence
    for (int idx = thread_id; idx < half_length; idx += CU1DBLOCK) {
        float2 val = reinterpret_cast<const float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }

    // Use shared memory and synchronize threads to handle special case efficiently
    __shared__ float shared_real;
    __shared__ float shared_im;

    if (threadIdx.x == 0) {
        shared_real = Ar[0];
        shared_im = Ar[row_length];
    }
    __syncthreads();

    // Update special case results
    if (threadIdx.x == 0) {
        Aw[0] = use_power ? shared_real * shared_real : fabs(shared_real);
        Aw[half_length] = use_power ? shared_im * shared_im : fabs(shared_im);
    }
}