#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy( int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    // Use shared memory for lifter coefficients to reduce global memory access
    __shared__ float shared_lifter_coeffs[1024];
    if (thread_id < num_cols) {
        shared_lifter_coeffs[thread_id] = lifter_coeffs[thread_id];
    }
    __syncthreads();

    float *feats = features + frame * ldf;

    // Apply lifter coefficients
    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += blockDim.x) { // Use blockDim.x instead of CU1DBLOCK for cleaner code
            float lift = shared_lifter_coeffs[c]; 
            float f = feats[c];
            feats[c] = f * lift;
        }
    }

    // Use the first thread in each block to apply energy
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = log(energy_floor);

        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        feats[0] = energy;
    }
}