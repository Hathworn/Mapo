#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Calculate the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for better memory access pattern
    for (int index = tid; index < N; index += blockDim.x * gridDim.x) {
        out[index] = in[index] > 0 ? in[index] : 0;
    }
}