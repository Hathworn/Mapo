#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for better readability.
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Check if the thread index is 0 to avoid unnecessary operations in parallel block.
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}