#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0;

    // Optimized reduction loop
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Use warp-level reduction for initial part
    if (blockDim.x <= warpSize) {
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            tmp += reduction_buffer[idx + offset];
        }
    } else {
        if (idx < warpSize) {
            tmp = reduction_buffer[idx];
            for (int i = warpSize; i < blockDim.x; i += warpSize) {
                tmp += reduction_buffer[i + idx];
            }
            reduction_buffer[idx] = tmp;
        }
    }
    __syncthreads();

    if (idx == 0) {
        for (int i = 1; i < min(blockDim.x, warpSize); ++i) {
            tmp += reduction_buffer[i];
        }
        reduction_buffer[0] = tmp;
    }
    __syncthreads();

    // Compute gradient
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}