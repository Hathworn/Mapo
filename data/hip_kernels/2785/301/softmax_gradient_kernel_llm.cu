#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    
    // Initialize tmp and compute partial dot product using warp-level parallelism
    float tmp = 0;
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Perform parallel reduction within the block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    // Compute gradient using reduced result
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}