#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogram_privatized_kernel(unsigned char *input, unsigned int *bins, unsigned int num_elements, unsigned int num_bins) {
    const int bx = blockIdx.x;
    const int bdx = blockDim.x;
    const int tx = threadIdx.x;
    const int gdx = gridDim.x;
    unsigned int tid = bx * bdx + tx;

    extern __shared__ unsigned int histo_s[]; // size is 3rd arg in <<< >>> of kernel
    // Initialize shared memory using block-level parallelism
    for (unsigned int bin_idx = tx; bin_idx < num_bins; bin_idx += bdx) {
        histo_s[bin_idx] = 0;
    }
    __syncthreads();

    const int bin_size = (num_elements - 1) / num_bins + 1;
    // Process elements using grid-stride loop
    for (unsigned int i = tid; i < num_elements; i += bdx * gdx) {
        int c = input[i] - 'a';
        if (c >= 0 && c < 26)
            atomicAdd(&(histo_s[c / bin_size]), 1);
    }
    __syncthreads();

    // Accumulate results to global memory
    for (unsigned int bin_idx = tx; bin_idx < num_bins; bin_idx += bdx) {
        atomicAdd(&(bins[bin_idx]), histo_s[bin_idx]);
    }
}