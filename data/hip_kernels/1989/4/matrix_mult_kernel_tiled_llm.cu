#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void matrix_mult_kernel_tiled(int *d_m, int *d_n, int *d_p, int m, int n, int k) {
    __shared__ int ds_m[TILE_WIDTH][TILE_WIDTH];
    __shared__ int ds_n[TILE_WIDTH][TILE_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    int pvalue = 0;

    for (int i = 0; i < (k + TILE_WIDTH - 1) / TILE_WIDTH; ++i) {
        // Load shared memory with bounds checks
        if (row < m && i * TILE_WIDTH + tx < k) 
            ds_m[ty][tx] = d_m[row * k + i * TILE_WIDTH + tx];
        else 
            ds_m[ty][tx] = 0;
        
        if (col < n && i * TILE_WIDTH + ty < k) 
            ds_n[ty][tx] = d_n[(i * TILE_WIDTH + ty) * n + col];
        else 
            ds_n[ty][tx] = 0;

        __syncthreads(); // Ensure loading is complete

        #pragma unroll
        for (int j = 0; j < TILE_WIDTH; j++) 
            pvalue += ds_m[ty][j] * ds_n[j][tx];

        __syncthreads(); // Prepare for next iteration
    }

    // Write result to global memory
    if (row < m && col < n) 
        d_p[row * n + col] = pvalue;
}