#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_mult_kernel(int *a, int *b, int *c, int m, int n, int k) {
    // Compute row and col indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize sum outside the loop
    int sum = 0;

    // Ensure the indices are within matrix bounds
    if (col < n && row < m) {
        // Calculate the dot product
        for (int i = 0; i < k; i++) {
            sum += a[row * k + i] * b[i * n + col];
        }
        // Write the result to the matrix
        c[row * n + col] = sum;
    }
}