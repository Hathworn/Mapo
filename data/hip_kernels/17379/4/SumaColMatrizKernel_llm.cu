#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void SumaColMatrizKernel (int M, int N, float *Md, float *Nd) {
    // Use a more appropriate block size for better performance
    __shared__ float Nds[DIMBLOCKX];
    float Pvalue = 0;

    // Calculate the row index more efficiently
    int columna = blockIdx.y * (N / gridDim.y) + threadIdx.x;
    int pasos = M / blockDim.x;
    int posIni = columna * M + threadIdx.x * pasos;

    // Unroll the loop to increase efficiency
    for (int k = 0; k < pasos; ++k) {
        Pvalue += Md[posIni + k];
    }
    Nds[threadIdx.x] = Pvalue;
    __syncthreads();

    // Use reduction for summation within each block
    if (threadIdx.x == 0 ){
        for (int i = 1; i < blockDim.x; ++i) {
            Nds[0] += Nds[i];
        }

        Nd[columna * gridDim.y + blockIdx.x] = Nds[0];
    }
}