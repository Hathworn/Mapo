#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VectorAdd(int *a, int *r, int n, double gamma)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < n)
    {
        double normalized_a = (double)a[i] / 255.0; // Normalize input value
        r[i] = (int)(255.0 * pow(normalized_a, 1.0 / gamma)); // Compute result using power function
    }
}