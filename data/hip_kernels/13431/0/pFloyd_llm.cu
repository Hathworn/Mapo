#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define INF 100000000
#define BLOCKSIZE 128
#define BITSFT 7 // log2(BLOCKSIZE)

__global__ void pFloyd(float *D, float *Q, int vertices, int k, int k2)
{
    int i = blockIdx.x;
    int j = (blockIdx.y << BITSFT) + threadIdx.x;
    int index = (i << vertices) + j;
    
    // Load common data into registers for efficiency
    float Dik = D[(i << vertices) + k];
    float Dkj = D[k2 + j];
    float Dij = D[index];

    // Use temporary variable to store updated distance
    float newDist = Dik + Dkj;

    // Compare and update directly
    if (newDist < Dij) {
        D[index] = newDist;
        Q[index] = k;
    }
}