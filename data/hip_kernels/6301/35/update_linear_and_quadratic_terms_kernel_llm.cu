#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    // Load cur_weight to a register
    float cur_weight = *cur_tot_weight;
    float val = 1.0f;

    // Avoid redundant type casting
    if (max_count > 0.0f) {
        float new_scale = fmaxf(cur_weight, (float)max_count) / max_count;
        val += (new_scale - 1.0f);
    }

    // Use thread indices to avoid branch divergence in the loop
    for (int32_t i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        quadratic[diag_idx] += val;
    }

    // Use warp divergence to update linear
    if (threadIdx.x == 0) {
        atomicAdd(&linear[0], val * prior_offset);
    }
}