#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;
    float *feats = features + frame * ldf;

    // Apply lifter coefficients with coalesced memory access
    if (cepstral_lifter != 0.0f) {
        int num_threads = blockDim.x;
        for (int c = thread_id; c < num_cols; c += num_threads) {
            feats[c] *= lifter_coeffs[c];
        }
    }

    // Use shared memory to store log_energy_floor to minimize recomputation
    __shared__ float log_energy_floor_shared;
    
    if (thread_id == 0) {
        log_energy_floor_shared = log(energy_floor);
    }
    __syncthreads();

    // Thread 0 for each frame will apply energy efficiently
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        if (energy_floor > 0.0f && energy < log_energy_floor_shared) {
            energy = log_energy_floor_shared;
        }
        feats[0] = energy;
    }
}