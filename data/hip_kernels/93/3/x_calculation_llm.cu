#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 256  // Define a constant block size for better occupancy

char* concat(char *s1, char *s2);

__global__ void x_calculation(float *x, float *r, float *r_squared, int size) {
    // Use shared memory to minimize global memory access
    __shared__ float alpha_shared;
    if (threadIdx.x == 0) {
        alpha_shared = r_squared[0];
    }
    __syncthreads();

    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < size) {
        x[index] += alpha_shared * r[index];
    }
}