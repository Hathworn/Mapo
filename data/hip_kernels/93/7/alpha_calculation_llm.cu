#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void alpha_calculation(float * r_squared ,float * p_sum, float * alpha)
{
    // Using local variables to minimize memory access
    float r_sq = r_squared[0];
    float p_s = p_sum[0];
    alpha[0] = r_sq / p_s;
}