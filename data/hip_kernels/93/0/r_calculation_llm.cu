#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

char* concat(char *s1, char *s2);

__global__ void r_calculation(float* a, int* indeces, float* b, float* x, float* r, int size)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    if (index < size)
    {
        // Use shared memory for performance improvement
        __shared__ float shared_a[3], shared_x[3];
        float sum = 0;

        #pragma unroll // Unroll loop to reduce overhead
        for (int i = 0; i < 3; i++)
        {
            // Load data into shared memory
            int idx = 3 * index + i;
            shared_a[i] = a[idx];
            shared_x[i] = x[indeces[idx]];
            __syncthreads(); // Ensure all threads have loaded the data

            sum += shared_a[i] * shared_x[i];
        }

        r[index] = b[index] - sum;
    }
}