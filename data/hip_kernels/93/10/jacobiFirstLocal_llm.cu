#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void jacobiFirstLocal(float *x, const float *diagonal_values, const float *non_diagonal_values, const int *indeces, const float *y, const int size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    float error = 1;
    float current_value = 1;

    if (index < size)
    {
        float local_diagonal_value = diagonal_values[index];
        float local_non_diagonal_values[2] = {
            non_diagonal_values[2 * index],
            non_diagonal_values[2 * index + 1]
        };
        
        int local_indeces[2] = {
            indeces[2 * index],
            indeces[2 * index + 1]
        };
        
        float local_y = y[index];
        
        while (fabsf(error) > 0.00001) // Simplify error check condition
        {
            float sum = local_non_diagonal_values[0] * x[local_indeces[0]] +
                        local_non_diagonal_values[1] * x[local_indeces[1]];

            current_value = (local_y - sum) / local_diagonal_value;
            error = current_value - x[index];
            x[index] = current_value;

            __syncthreads(); // Ensure all threads are in sync for current iteration
        }
    }
}