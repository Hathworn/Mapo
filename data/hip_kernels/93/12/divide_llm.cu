#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void divide(float *x, float* y, float* out, const int size)
{
    // Calculate the global index
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index is within range
    if (index < size)
    {
        // Perform the division
        out[index] = x[index] / y[index];
    }
}