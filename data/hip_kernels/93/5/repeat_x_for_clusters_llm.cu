#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void repeat_x_for_clusters(float *x, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within size directly, no need for block index check
    if (index < size)
    {
        // Use shared memory to reduce redundant global memory access
        __shared__ float shared_mem[blockDim.x];
        shared_mem[threadIdx.x] = x[threadIdx.x];
        __syncthreads();

        x[index] = shared_mem[threadIdx.x];
    }
}