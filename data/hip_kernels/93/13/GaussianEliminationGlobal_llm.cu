#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GaussianEliminationGlobal(const int clusterSize, float *x, const float *diagonal_values, const float *non_diagonal_values, float *y, const int size)
{
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int gi = index * clusterSize;
    // Optimize: Use shared memory for the cluster matrix within a block
    __shared__ float matrix[180][180];

    // Initialize matrix with zeros and set diagonal values
    for (int i = 0; i < clusterSize; ++i) {
        for (int j = 0; j < clusterSize; ++j) {
            matrix[i][j] = 0.0f;
        }
        matrix[i][i] = diagonal_values[gi + i];
    }

    // Set non-diagonal values
    for (int i = 0; i < clusterSize - 1; ++i) {
        matrix[i][i + 1] = non_diagonal_values[2 * (gi + i) + 1];
        matrix[i + 1][i] = non_diagonal_values[2 * (gi + i) + 2];
    }

    // Triangle form (Gaussian elimination)
    for (int i = 0; i < clusterSize; ++i) {
        for (int j = i + 1; j < clusterSize; ++j) {
            float ratio = matrix[j][i] / matrix[i][i];
            for (int k = 0; k < clusterSize; ++k) {
                matrix[j][k] -= matrix[i][k] * ratio;
            }
            y[gi + j] -= y[gi + i] * ratio;
        }
    }

    __syncthreads();

    // Back substitution
    for (int i = clusterSize - 1; i >= 0; --i) {
        float current = 0.0f;
        for (int j = i; j < clusterSize; ++j) {
            current += matrix[i][j] * x[gi + j];
        }
        x[gi + i] = (y[gi + i] - current) / matrix[i][i];
    }
}