#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ap_multiplication(float *values, int *indeces, float *r, float *p_sum, int size) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    // Check bounds before accessing shared memory
    if (index < size) {
        float sum = 0.0f; // Use local variable for accumulation
        for (int i = 0; i < 3; i++) {
            sum += values[3 * index + i] * r[indeces[3 * index + i]];
        }
        p_sum[index] = sum; // Write back to global memory
    }
}