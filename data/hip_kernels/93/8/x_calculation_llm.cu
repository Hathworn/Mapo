#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void x_calculation(float *x, float *r, float *r_squared, int size)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < size)
    {
        // Cache r_squared[0] to reduce memory access
        float alpha = __ldg(&r_squared[0]);
        x[index] += alpha * r[index];
    }
}