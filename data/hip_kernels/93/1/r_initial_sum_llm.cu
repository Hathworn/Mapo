#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

char* concat(char *s1, char *s2);

__global__ void r_initial_sum(float* a, int* indeces, float* x, float* r, float* r_squared, float* p_sum, int size)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int local_index = threadIdx.x;

    // Use dynamic shared memory to avoid hardcoded size
    extern __shared__ float shared_memory[];

    float* shared_r_squared = shared_memory;
    float* shared_p_sum = shared_memory + blockDim.x;

    shared_r_squared[local_index] = 0;
    shared_p_sum[local_index] = 0;
    __syncthreads();

    if (index < size)
    {
        // Unrolling loop for performance optimization
        #pragma unroll
        for (int i = 0; i < 3; i++)
        {
            shared_p_sum[local_index] += a[3 * index + i] * r[indeces[3 * index + i]];
        }

        shared_r_squared[local_index] = r[index] * r[index];
        shared_p_sum[local_index] *= r[index];
    }
    __syncthreads();

    // Optimize reduction using warp and shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (local_index < s)
        {
            shared_r_squared[local_index] += shared_r_squared[local_index + s];
            shared_p_sum[local_index] += shared_p_sum[local_index + s];
        }
        // Separate __syncthreads outside if condition in the loop
        __syncthreads();
    }

    if (local_index == 0)
    {
        r_squared[blockIdx.x] = shared_r_squared[0];
        p_sum[blockIdx.x] = shared_p_sum[0];
    }
}