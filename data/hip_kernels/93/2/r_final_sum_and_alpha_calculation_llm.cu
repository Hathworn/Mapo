#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

char* concat(char *s1, char *s2);

__global__ void r_final_sum_and_alpha_calculation(float * r_squared ,float * p_sum ,int size)
{
    int index = threadIdx.x;
    
    // Use dynamically allocated shared memory to handle variable sizes
    extern __shared__ float shared_memory[];
    float* shared_r_squared = shared_memory;
    float* shared_p_sum = shared_memory + blockDim.x;
    
    if (index < size) {
        // Load data into shared memory
        shared_r_squared[index] = r_squared[index];
        shared_p_sum[index] = p_sum[index];
    } else {
        shared_r_squared[index] = 0;
        shared_p_sum[index] = 0;
    }
    __syncthreads();
    
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (index < s) {
            // Perform reduction in shared memory using warp synchronous operations
            shared_r_squared[index] += shared_r_squared[index + s];
            shared_p_sum[index] += shared_p_sum[index + s];
        }
        __syncthreads();
    }
    
    if (index == 0) {
        // Calculate alpha only once for each block
        r_squared[blockIdx.x] = shared_r_squared[0] / shared_p_sum[0];
    }
}