#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GaussianEliminationShared(const int clusterSize, float *x, const float *diagonal_values, const float *non_diagonal_values, float *y)
{
    const int index = blockIdx.x;

    // Use dynamic shared memory allocation
    extern __shared__ float shared_m[];

    // Initialize shared memory
    for (int i = threadIdx.x; i < clusterSize * clusterSize; i += blockDim.x)
    {
        shared_m[i] = 0;
    }
    __syncthreads();

    // Load diagonal values
    for (int i = threadIdx.x; i < clusterSize; i += blockDim.x)
    {
        shared_m[i * clusterSize + i] = diagonal_values[clusterSize * index + i];
    }

    // Load non-diagonal values
    for (int i = threadIdx.x; i < clusterSize - 1; i += blockDim.x)
    {
        shared_m[i * clusterSize + i + 1] = non_diagonal_values[clusterSize * index * 2 + 2 * i + 1];
        shared_m[(i + 1) * clusterSize + i] = non_diagonal_values[clusterSize * index * 2 + 2 * i + 2];
    }
    __syncthreads();

    // Triangle form
    for (int i = 0; i < clusterSize; ++i)
    {
        if (threadIdx.x == 0)
        {
            for (int j = i + 1; j < clusterSize; ++j)
            {
                double ratio = shared_m[j * clusterSize + i] / shared_m[i * clusterSize + i];
                for (int k = 0; k < clusterSize; ++k)
                {
                    shared_m[j * clusterSize + k] -= shared_m[i * clusterSize + k] * ratio;
                }
                y[clusterSize * index + j] -= y[clusterSize * index + i] * ratio;
            }
        }
        __syncthreads();
    }

    // Back substitution
    for (int i = clusterSize - 1; i > -1; --i)
    {
        double current = 0;
        for (unsigned int j = i + 1; j < clusterSize; ++j)
        {
            current += shared_m[i * clusterSize + j] * x[clusterSize * index + j];
        }
        if (threadIdx.x == 0)
        {
            x[clusterSize * index + i] = (y[clusterSize * index + i] - current) / shared_m[i * clusterSize + i];
        }
        __syncthreads();
    }
}