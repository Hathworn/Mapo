#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cg_zero_start(float* a, float* x, float* b, int size)
{
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int local_index = threadIdx.x;

    __shared__ float shared_r_squared[1024];
    __shared__ float shared_p_sum[1024];
    float local_b;

    // Initialize shared memory
    shared_r_squared[local_index] = 0;
    shared_p_sum[local_index] = 0;
    
    __syncthreads();

    if (index < size)
    {
        local_b = b[index + 2 * blockIdx.x + 1]; // Reduce usage of blockIdx.x

        // Compute shared_p_sum
        for (int i = 0; i < 3; i++)
        {
            shared_p_sum[local_index] += a[3 * index + i] * b[index + 2 * blockIdx.x + i];
        }

        __syncthreads();

        shared_r_squared[local_index] = local_b * local_b;
        shared_p_sum[local_index] *= local_b;

        __syncthreads();

        // Reduce using a single thread synchronization
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
        {
            if (local_index < s)
            {
                shared_r_squared[local_index] += shared_r_squared[local_index + s];
                shared_p_sum[local_index] += shared_p_sum[local_index + s];
            }
            __syncthreads();
        }

        // Compute alpha and update x using first thread of the block
        if (local_index == 0)
        {
            float alpha = shared_r_squared[0] / shared_p_sum[0];
            // Use atomic operation to avoid race condition
            atomicAdd(&x[index], alpha * local_b);
        }
    }
}