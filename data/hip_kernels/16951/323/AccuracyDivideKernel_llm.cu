#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory to reduce global memory access latency
    __shared__ float shared_accuracy;
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy;
    }
    __syncthreads();
    
    // Divide once per block, then write back to global memory
    if (threadIdx.x == 0) {
        shared_accuracy /= N;
        *accuracy = shared_accuracy;
    }
}