#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

struct pixel //to store RGB values
{
    unsigned char r;
    unsigned char g;
    unsigned char b;
};

__device__ pixel padding(pixel* Pixel_val, int x_coord, int y_coord, int img_width, int img_height)
{
    pixel Px = {0, 0, 0}; // Initialize pixel to zero
    if (x_coord < img_width && y_coord < img_height && x_coord >= 0 && y_coord >= 0)
    {
        Px = Pixel_val[y_coord * img_width + x_coord];
    }
    return Px;
}

__global__ void horizontal_conv(pixel* Pixel_in, pixel* Pixel_out, int img_wd, int img_ht, float* kernel, int k)
{
    __shared__ float shared_kernel[256]; // Use shared memory for kernel

    size_t col = blockIdx.x * blockDim.x + threadIdx.x;
    size_t row = blockIdx.y * blockDim.y + threadIdx.y;
    size_t pix_idx = row * img_wd + col;

    if (threadIdx.x < k) {
        shared_kernel[threadIdx.x] = kernel[threadIdx.x]; // Load kernel into shared memory
    }
    __syncthreads(); // Ensure all threads have loaded the kernel

    float tmp_r = 0, tmp_g = 0, tmp_b = 0;
    if (row < img_ht && col < img_wd)
    {
        for (int l = 0; l < k; l++)
        {
            pixel pix_val = padding(Pixel_in, col + l - (k - 1) / 2, row, img_wd, img_ht);
            tmp_r += pix_val.r * shared_kernel[l];
            tmp_g += pix_val.g * shared_kernel[l];
            tmp_b += pix_val.b * shared_kernel[l];
        }
        Pixel_out[pix_idx].r = tmp_r;
        Pixel_out[pix_idx].g = tmp_g;
        Pixel_out[pix_idx].b = tmp_b;
    }
}