#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_iteration(char* buffer, char* out_buffer, size_t pitch, size_t pitch_out, int width, int height)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Early exit for threads outside the valid computation area
    if (x >= width || y >= height)
        return;

    // Using local variables for repetitive calculations to improve readability and performance
    int idx = y * pitch + x;
    int left_x = (x - 1 + width) % width;
    int right_x = (x + 1) % width;
    int up_y = (y - 1 + height) % height;
    int down_y = (y + 1) % height;

    // Optimize access by storing common expressions to reduce repeated calculations and improve memory coalescing
    int up_idx = up_y * pitch;
    int down_idx = down_y * pitch;

    char n_alive = buffer[up_idx + left_x] + buffer[up_idx + x]
                 + buffer[up_idx + right_x] + buffer[idx - 1]
                 + buffer[idx + 1] + buffer[down_idx + left_x]
                 + buffer[down_idx + x] + buffer[down_idx + right_x];

    // Write the result into the output buffer
    out_buffer[idx] = (n_alive == 3 || (buffer[idx] && n_alive == 2));
}