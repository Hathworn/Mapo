#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define PI 3.141592653
#define PREC 20
#define maxNeighbors 6

typedef double4 particle;
typedef double dbl;

__global__ void LEShift(particle *parts, dbl LEshear) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < blockDim.x * gridDim.x) {  // Ensure index is within bounds
        parts[i].y += parts[i].x * LEshear;  // Simplify operation
    }
}