#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_cmp_kernel(std::size_t n, int* aptr, int* bptr, int* rptr) {
  // Calculate global thread index 
  int i = threadIdx.x + blockIdx.x * blockDim.x;

  // Initialize shared memory to store comparison result
  __shared__ int result;

  if (threadIdx.x == 0) result = 0;

  __syncthreads();
  
  // Perform comparison, store result in shared memory
  if (i < n) {
      atomicOr(&result, aptr[i] < bptr[i]);
  }

  __syncthreads();

  // Write the final result to global memory from one thread
  if (threadIdx.x == 0 && result) {
      *rptr = 1;
  }
}