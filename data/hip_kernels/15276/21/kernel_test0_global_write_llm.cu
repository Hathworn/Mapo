#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    // Use local variables to minimize memory access latency
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int pattern = 1;
    unsigned long mask = 4;

    while (ptr < end_ptr) {
        unsigned int* next_ptr = (unsigned int*)(((unsigned long)ptr) | mask);
        
        // Check boundary condition before accessing memory
        if (next_ptr < ptr || next_ptr >= end_ptr) {
            mask = mask << 1;
            continue;
        }

        *next_ptr = pattern;
        
        // Update pattern and mask efficiently
        pattern = pattern << 1;
        mask = mask << 1;
        ptr = next_ptr;
    }
}