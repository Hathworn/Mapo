#include "hip/hip_runtime.h"
```cuda
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test5_init(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);

    // Check bounds early to avoid unnecessary computation
    if (ptr >= (unsigned int*) end_ptr) return;

    unsigned int p1 = 1;
    for (unsigned int i = 0; i < BLOCKSIZE/sizeof(unsigned int); i+=16) {
        unsigned int p2 = ~p1;

        // Unrolled loop for performance
        ptr[i] = p1;
        ptr[i+1] = p1;
        ptr[i+2] = p2;
        ptr[i+3] = p2;
        ptr[i+4] = p1;
        ptr[i+5] = p1;
        ptr[i+6] = p2;
        ptr[i+7] = p2;
        ptr[i+8] = p1;
        ptr[i+9] = p1;
        ptr[i+10] = p2;
        ptr[i+11] = p2;
        ptr[i+12] = p1;
        ptr[i+13] = p1;
        ptr[i+14] = p2;
        ptr[i+15] = p2;

        // Update p1 efficiently
        p1 <<= 1;
        if (p1 == 0) p1 = 1;
    }
}