#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early exit if block out of bounds
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }
    
    // Calculate the number of elements to process
    unsigned int num_elements = BLOCKSIZE / sizeof(unsigned long);

    // Calculate the start index for this thread
    unsigned int start_index = threadIdx.x;
    
    // Iterate over the array using a stride equal to blockDim.x
    for (unsigned int i = start_index; i < num_elements; i += blockDim.x) {
        ptr[i] = (unsigned long) &ptr[i];
    }
}