#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);

    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Use threadIdx to improve parallelism and reduce loop iterations
    unsigned int idx = threadIdx.x;
    unsigned int stride = blockDim.x;
    
    // Each thread writes multiple elements according to step size
    for (unsigned int i = idx; i < BLOCKSIZE / sizeof(unsigned int); i += stride) {
        ptr[i] = pattern;
    }
}