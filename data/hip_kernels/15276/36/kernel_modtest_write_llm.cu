#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    // Calculate thread-specific starting pointer
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check if the pointer exceeds the boundary
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Loop unrolling to optimize memory writes
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE/sizeof(unsigned int); i++){
        if (i % MOD_SZ == offset) {
            ptr[i] = p1;  // Assign `p1` when condition met
        } else {
            ptr[i] = p2;  // Assign `p2` otherwise
        }
    }
}