#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    // Calculate the initial pointer position for this block
    unsigned int* orig_ptr = reinterpret_cast<unsigned int*>(_ptr) + blockIdx.x * BLOCKSIZE / sizeof(unsigned int);
    unsigned int* ptr = orig_ptr;

    // Check if the initial pointer is already out of bounds
    if (ptr >= reinterpret_cast<unsigned int*>(end_ptr)) {
        return;
    }

    // Define the ending position of the block
    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);

    // Initialize pattern and mask
    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Set the initial value
    *ptr = pattern;

    // Optimized loop with reduced computations and more efficient pointer management
    while (true) {
        ptr = reinterpret_cast<unsigned int*>(reinterpret_cast<unsigned long>(orig_ptr) | mask);

        // Break conditions
        if (ptr == orig_ptr || ptr >= block_end) {
            break;
        }

        // Update the memory at the identified pointer location
        *ptr = pattern;

        // Update the pattern and mask for the next iteration
        pattern <<= 1;
        mask <<= 1;
    }
}