#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check if pointer exceeds end_ptr
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Use parallel threads to write pattern to memory
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        ptr[i] = pattern;
    }
}