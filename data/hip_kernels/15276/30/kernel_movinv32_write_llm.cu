#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    // Calculate the thread's unique global offset
    unsigned int* ptr = (unsigned int*) (_ptr + (blockIdx.x * BLOCKSIZE + threadIdx.x * sizeof(unsigned int)));
    
    // Check if the pointer is out of bounds, return early
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }
    
    // Using shared memory to reduce global memory access latency
    __shared__ unsigned int shared_pattern[BLOCKSIZE / sizeof(unsigned int)];
    
    // Initialize pattern variables
    unsigned int k = offset;
    unsigned int pat = pattern;
    
    // Sequential processing replaced by parallel processing on threads within the block
    shared_pattern[threadIdx.x] = pat;
    k++;
    if (k >= 32) {
        k = 0;
        shared_pattern[threadIdx.x] = lb;
    } else {
        shared_pattern[threadIdx.x] = shared_pattern[threadIdx.x] << 1 | sval;
    }
    
    ptr[threadIdx.x] = shared_pattern[threadIdx.x];
}