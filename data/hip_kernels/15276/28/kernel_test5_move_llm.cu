#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check if pointer reaches or exceeds the end boundary
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Calculate half_count once
    unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Combine two loops to reduce kernel launch iterations
    for (unsigned int i = 0; i < half_count; i++) {
        if (i < half_count - 8) {
            ptr[i + 8] = ptr_mid[i];
        }
        if (i >= half_count - 8) {
            ptr[i - (half_count - 8)] = ptr_mid[i];
        }
    }

    return;
}