#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define _size 512

__global__ void mul(int *a, int *b, int *c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x; // Precompute the index for better readability and efficiency
    c[index] = a[index] * b[index]; // Use precomputed index for operations
}