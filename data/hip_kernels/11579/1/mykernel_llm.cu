#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mykernel(void) {
    // Combine printf statements for efficiency
    int global_id = threadIdx.x + blockIdx.x * blockDim.x; // Precompute global id
    printf("Block id: %d, Thread id: %d, Global id: %d\n", blockIdx.x, threadIdx.x, global_id);
}