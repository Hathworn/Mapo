#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    const float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;

    // Unroll loop for better efficiency
    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        if (idx == 0) continue; // Skip index 0 as it's a special case

        float2 val = reinterpret_cast<const float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret); // Ternary operator for concise condition
    }

    // Handle special case with thread synchronization
    __syncthreads();
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        Aw[0] = use_power ? real * real : fabs(real); // Ternary operator for concise condition
        Aw[half_length] = use_power ? im * im : fabs(im); // Ternary operator for concise condition
    }
}