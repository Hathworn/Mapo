#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    float *Ar = A_in + blockIdx.x * ldi;
    float *Aw = A_out + blockIdx.x * ldo;

    int half_length = row_length / 2;
    
    // Use stride loop to handle more work per thread
    for (int idx = thread_id; idx < half_length; idx += stride) {
        // Process calculation only if idx is not zero
        if (idx > 0) {
            float2 val = reinterpret_cast<float2 *>(Ar)[idx];
            float ret = val.x * val.x + val.y * val.y;
            Aw[idx] = use_power ? ret : sqrtf(ret);
        }
    }

    // Handle special case for the first thread in each block
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}