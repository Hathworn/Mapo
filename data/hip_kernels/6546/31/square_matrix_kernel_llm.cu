#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Calculate row and column index for each thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure that thread indices are within matrix bounds
    if(row < num_rows && col < num_cols) {
        // Directly calculate and store the squared value
        float f = feats[row * ldf + col];
        feats_sq[row * lds + col] = f * f;
    }
}