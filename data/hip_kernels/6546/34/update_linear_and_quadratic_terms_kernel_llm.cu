#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float old_num_frames, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    // Load current weight once into a register to minimize global memory access
    float cur_weight = *cur_tot_weight;

    float new_num_frames = old_num_frames + cur_weight;
    float prior_scale_change = 1.0f;

    if (max_count != 0) {
        float old_prior_scale = max(old_num_frames, (float)max_count) / max_count;
        float new_prior_scale = max(new_num_frames, (float)max_count) / max_count;
        prior_scale_change += new_prior_scale - old_prior_scale;
    }

    // Optimize thread calculation by reducing unnecessary arithmetic
    int32_t i_start = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t i_stride = blockDim.x * gridDim.x;
    for (int32_t i = i_start; i < n; i += i_stride) {
        int32_t diag_idx = ((i + 1) * (i + 2) / 2) - 1;
        quadratic[diag_idx] += prior_scale_change;
    }

    // Use atomic operation for potential data race; only first thread updates
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        atomicAdd(&linear[0], prior_offset * prior_scale_change);
    }
}