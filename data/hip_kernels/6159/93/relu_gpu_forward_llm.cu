#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, const float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Unrolling for better performance, assuming blockDim.x is a multiple of 4
    if (tid < N) {
        out[tid] = in[tid] > 0 ? in[tid] : 0;
    }

    if (tid + blockDim.x < N) {
        out[tid + blockDim.x] = in[tid + blockDim.x] > 0 ? in[tid + blockDim.x] : 0;
    }

    if (tid + 2 * blockDim.x < N) {
        out[tid + 2 * blockDim.x] = in[tid + 2 * blockDim.x] > 0 ? in[tid + 2 * blockDim.x] : 0;
    }

    if (tid + 3 * blockDim.x < N) {
        out[tid + 3 * blockDim.x] = in[tid + 3 * blockDim.x] > 0 ? in[tid + 3 * blockDim.x] : 0;
    }
}