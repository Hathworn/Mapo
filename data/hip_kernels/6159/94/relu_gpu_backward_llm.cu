#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        // Use ternary for direct assignment to optimize control flow
        ingrad[tid] = (indata[tid] > 0) * outgrad[tid];
    }
}