#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_from_gpu(void)
{
    int global_tid = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread ID
    printf("Hello World from block %d and thread %d!\n", blockIdx.x, global_tid);  // Use global thread ID
}