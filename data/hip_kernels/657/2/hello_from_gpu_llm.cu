#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_from_gpu(void)
{
    // Using blockIdx and threadIdx to specify unique identifiers in the output
    printf("Hello World from the GPU! Block %d, Thread %d\n", blockIdx.x, threadIdx.x);
}