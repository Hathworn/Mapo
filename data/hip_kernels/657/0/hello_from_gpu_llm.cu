#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello_from_gpu(void)
{
    // Use threadIdx.x or blockIdx.x for more dynamic message printing
    const int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx == 0) 
    {
        // Print message from only one thread to avoid redundant output
        printf("Hello World from the GPU!\n");
    }
}