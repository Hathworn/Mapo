#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dev_const(float *px, float k) {
    // Compute global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate total number of threads
    int total_threads = gridDim.x * blockDim.x;

    // Loop over elements in steps of total_threads
    for(int i = tid; i < total_threads; i += total_threads) {
        px[i] = k;
    }
}