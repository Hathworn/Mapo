#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

}

__global__ void vsub(const float *a, const float *b, float *c, int n)
{
    // Compute thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure within bounds
    if (i < n) {
        c[i] = a[i] - b[i];
    }
}