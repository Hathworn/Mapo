#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

}
__global__ void vdiv(const float *a, const float *b, float *c, const int array_size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Bounds check to prevent accessing beyond array limits
    if (i < array_size) {
        c[i] = a[i] / b[i];
    }
}