#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Fprop1(const float* in, const float* syn1, float* layer1)
{
    int i = threadIdx.x;                         // 256
    int k = blockIdx.x;                          // Data.count

    // Using shared memory to optimize access
    extern __shared__ float shared_syn1[];
    
    // Load syn1 into shared memory
    for (int j = threadIdx.x; j < 28*28; j += blockDim.x) {
        shared_syn1[j] = syn1[j*256 + i];
    }
    __syncthreads();

    float x = 0.0;
    for (int j = 0; j < 28*28; ++j) {
        x += in[k*28*28 + j] * shared_syn1[j];
    }
    layer1[k*256 + i] = x;
}