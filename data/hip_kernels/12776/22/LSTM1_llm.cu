#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float tanh_(float x)
{
    float exp2x = exp(2.0 * x);
    return (exp2x - 1.0) / (exp2x + 1.0);
}

__global__ void LSTM1(float* __restrict__ layer1, float* __restrict__ lstm1, const float* __restrict__ gate1i, const float* __restrict__ gate1o, const int offset)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x; // 256 threads per block

    // Use registers to minimize redundant memory access
    float g_i = gate1i[256 * offset + i];
    float g_f = 1.0f - g_i;
    float g_o = gate1o[256 * offset + i];
    float layer1_val = layer1[256 * offset + i];
    
    float i_t = tanh_(layer1_val) * g_i;
    float i_p = 0.0f;

    if (offset > 0) {
        i_p = g_f * lstm1[256 * (offset - 1) + i];
    }

    float sum = i_p + i_t;
    lstm1[256 * offset + i] = sum;
    layer1[256 * offset + i] = tanh_(sum) * g_o;
}