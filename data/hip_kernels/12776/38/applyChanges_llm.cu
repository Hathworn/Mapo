#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void applyChanges(float* syn, float* dsyn, int dim, float alpha)
{
    // Calculate unique thread index within entire grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Ensure all threads cover the memory updates
    for (int j = idx; j < dim*dim; j += stride) {
        syn[j] += dsyn[j] * alpha;
    }
}