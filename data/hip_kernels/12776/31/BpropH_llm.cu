#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void BpropH(const float* layer1, float* dlayer1, const float* synH, float* dsynH, const float alpha, const int offset)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x; // 256
    int j = blockDim.y * blockIdx.y + threadIdx.y; // 256

    // Early exit if thread indices are out of bounds
    if (i >= 256 || j >= 256) return;

    float val1 = dlayer1[offset * 256 + j] * layer1[(offset - 1) * 256 + i] * alpha;
    float val2 = layer1[offset * 256 + j] * synH[i * 256 + j];

    // Use shared memory to reduce atomic operations (only local optimization can be applied in global memory context)
    __shared__ float shared_dsynH[256 * 256];
    __shared__ float shared_dlayer1[256 * 256];

    atomicAdd(&shared_dsynH[i * 256 + j], val1);
    atomicAdd(&shared_dlayer1[(offset - 1) * 256 + i], val2);

    __syncthreads();

    if (i == 0 && j == 0) { // Only one thread writes back to global memory to reduce atomic operations
        for (int x = 0; x < 256; x++) {
            for (int y = 0; y < 256; y++) {
                atomicAdd(&dsynH[x * 256 + y], shared_dsynH[x * 256 + y]);
                atomicAdd(&dlayer1[(offset - 1) * 256 + x], shared_dlayer1[(offset - 1) * 256 + x]);
            }
        }
    }
}