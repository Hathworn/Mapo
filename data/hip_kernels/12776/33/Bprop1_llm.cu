#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Bprop1(const float* dlayer1, const float* dlayer1i, const float* dlayer1o, const float* in, float* dsyn1, float* dsyn1i, float* dsyn1o, const float alpha)
{
    int i = blockDim.y * blockIdx.y + threadIdx.y; // 64
    int j = threadIdx.x;                           // 256
    int k = blockIdx.x;                            // Data.count

    // Cache value for optimization
    float in_cache = in[k * 64 + i] * alpha;

    // Separate indices calculations to reduce redundant computations
    int offset = i * 256 + j;
    int dlayer_offset = k * 256 + j;

    // Use local variable to minimize memory latency
    float contrib_dsyn1  = dlayer1[dlayer_offset] * in_cache;
    float contrib_dsyn1i = dlayer1i[dlayer_offset] * in_cache;
    float contrib_dsyn1o = dlayer1o[dlayer_offset] * in_cache;

    // Perform atomic additions
    atomicAdd(&dsyn1[offset],  contrib_dsyn1);
    atomicAdd(&dsyn1i[offset], contrib_dsyn1i);
    atomicAdd(&dsyn1o[offset], contrib_dsyn1o);
}