#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forwardPass1(float* in, float* syn1, float* layer1)
{
    int l = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    int Y = 128;

    // Use a shared memory to reduce global memory accesses
    extern __shared__ float s_layer1[];
    if (threadIdx.y == 0)
        s_layer1[threadIdx.x] = 0.0f;
    __syncthreads();

    // Use atomicAdd on shared memory
    atomicAdd(&s_layer1[threadIdx.x], in[j] * syn1[j * Y + l]);
    __syncthreads();

    // Write results back to global memory
    if (threadIdx.y == 0) {
        layer1[l] = s_layer1[threadIdx.x];
        layer1[l] = 1.0f / (1.0f + exp(-layer1[l]));  // Apply sigmoid activation
    }
}