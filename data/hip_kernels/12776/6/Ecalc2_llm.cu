#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Ecalc2(float* __restrict__ out, const float* __restrict__ label)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < gridDim.x * blockDim.x) { // Check bounds
        out[i] = label[i] - out[i];
    }
}