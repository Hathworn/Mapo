#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Fprop1(const float* in, const float* syn1, float* layer1)
{
    int i = threadIdx.x;                         // 256
    int k = blockIdx.x;                          // Data.count
    float x = 0.0;

    int stride = blockDim.x;                     // Optimize with loop unrolling
    for (int j = 0; j < 28*28; j += stride) {    
        int idx = j * 256 + i;
        float in_val = in[k * 28 * 28 + j + i];
        float syn_val = syn1[idx];
        x += in_val * syn_val;

        // Unroll the loop
        if ((j + 1) < 28 * 28) {
            in_val = in[k * 28 * 28 + j + i + 1];
            syn_val = syn1[idx + 256];
            x += in_val * syn_val;
        }
        if ((j + 2) < 28 * 28) {
            in_val = in[k * 28 * 28 + j + i + 2];
            syn_val = syn1[idx + 512];
            x += in_val * syn_val;
        }
        if ((j + 3) < 28 * 28) {
            in_val = in[k * 28 * 28 + j + i + 3];
            syn_val = syn1[idx + 768];
            x += in_val * syn_val;
        }
    }
    layer1[k * 256 + i] = x;
}