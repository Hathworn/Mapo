#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Ecalc2(float* out, const float* label) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Flatten 2D grid to 1D index

    if (i < blockDim.x * gridDim.x)                // Ensure index is within bounds
    {
        out[i] = label[i] - out[i];
    }
}