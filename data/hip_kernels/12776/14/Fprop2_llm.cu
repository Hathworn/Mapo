#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Fprop2(const float* layer1, const float* syn2, float* out)
{
    int i = blockDim.y * blockIdx.y + threadIdx.y;  // 10
    int j = blockIdx.x;  // Data.count

    // Use shared memory to reduce global memory accesses
    __shared__ float sharedLayer1[256];
    float x = 0.0;

    for (int k = 0; k < 256; ++k)
    {
        sharedLayer1[k] = layer1[j * 256 + k];
        __syncthreads();  // Ensure all threads have written to shared memory

        x += sharedLayer1[k] * syn2[k * 10 + i];
    }

    out[j * 10 + i] = x;
}