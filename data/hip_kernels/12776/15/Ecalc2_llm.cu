#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Ecalc2(float* out, const float* label)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x; // Calculate global index
    if (idx < /* Add appropriate bounds checking logic, e.g., array size */) {
        out[idx] = label[idx] - out[idx]; // Perform subtraction operation
    }
}