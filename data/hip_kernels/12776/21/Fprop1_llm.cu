#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Fprop1(const float* in, const float* syn1, float* layer1)
{
    int i = threadIdx.x;                         
    int j = blockDim.y * blockIdx.y + threadIdx.y; 
    int k = blockIdx.x;                          

    // Allocate shared memory for partial results
    __shared__ float partialSum[256][64]; 
    partialSum[i][threadIdx.y] = in[64 * k + j] * syn1[j * 256 + i];
    __syncthreads();

    // Use a single thread to accumulate results for each output element
    if (threadIdx.y == 0) {
        float sum = 0.0f;
        for (int t = 0; t < blockDim.y; ++t) {
            sum += partialSum[i][t];
        }
        atomicAdd(&layer1[256 * k + i], sum);
    }
}