#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Bprop2(const float* out, const float* layer1, float* dsyn2, const int count, const float alpha)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y; // 256
    int j = blockIdx.x * blockDim.x + threadIdx.x; // 4
    
    // Load out[j] into a register
    float out_val = out[j];

    // Load layer1 value into a register
    float layer1_val = layer1[256*(count) + i];

    // Perform multiplication in a register
    float product = out_val * layer1_val * alpha;

    // Atomic addition to avoid race conditions
    atomicAdd(&dsyn2[i*4 + j], product);
}