#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Fprop2(const float* layer1, const float* syn2, float* out, const int offset)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x; // Compute global index
    int k = blockDim.y * blockIdx.y + threadIdx.y; // Compute global index
    float temp = layer1[256 * offset + k] * syn2[k * 4 + i]; // Calculate temporary result

    // Use shared memory to reduce contention on atomic operations
    __shared__ float sharedOut[1024]; // Adjust size according to your blockDim.x
    sharedOut[threadIdx.x] = 0.0;
    __syncthreads();

    atomicAdd(&sharedOut[threadIdx.x], temp); // Local atomic addition
    __syncthreads();

    atomicAdd(&out[i], sharedOut[threadIdx.x]); // Global atomic addition
}
```
