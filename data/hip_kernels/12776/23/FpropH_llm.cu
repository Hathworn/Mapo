#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void FpropH(float* layer1, const float* synH, const int offset)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Load data into shared memory to optimize memory access
    __shared__ float shared_layer1[256];
    if (threadIdx.y == 0) {
        shared_layer1[threadIdx.x] = layer1[256 * (offset - 1) + i];
    }
    __syncthreads();

    // Use shared memory for the calculation
    atomicAdd(&layer1[256 * offset + j], shared_layer1[i] * synH[i * 256 + j]);
}