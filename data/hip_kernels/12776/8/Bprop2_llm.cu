#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Utilize shared memory for improved access speed
__global__ void Bprop2(const float* layer1, float* dsyn2, const float* out, const float alpha)
{
    int i = threadIdx.x; // 256
    int j = blockDim.y * blockIdx.y + threadIdx.y; // 10
    int k = blockIdx.x;  // Data.count

    extern __shared__ float shared_layer1[];
    
    // Load data into shared memory to reduce global memory accesses
    shared_layer1[threadIdx.x] = layer1[256 * k + i];
    __syncthreads();

    atomicAdd(&dsyn2[i * 10 + j], out[k * 10 + j] * shared_layer1[threadIdx.x] * alpha);
}