#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Bprop1(const float* in, float* dsyn1, const float* dlayer1, const float alpha)
{
    // Calculate unique thread indices
    int i = blockDim.y * blockIdx.y + threadIdx.y; // 28*28
    int j = threadIdx.x;                           // 256
    int k = blockIdx.x;                            // Data.count

    // Use register variables to avoid re-computation and minimize memory access
    float dlayer1_val = dlayer1[k * 256 + j];
    float in_val = in[k * 28 * 28 + i];
    float result = dlayer1_val * in_val * alpha;

    // Atomic addition to handle concurrent writes by multiple threads
    atomicAdd(&dsyn1[i * 256 + j], result);
}