#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forwardPass2(float* layer1, float* syn2, float* out)
{
    int l = blockDim.x * blockIdx.x + threadIdx.x;
    int Y = 128;
    int Z = 10;

    // Initialize the output element before accumulation
    float sum = 0.0f;

    // Use a register for the current index calculation
    #pragma unroll
    for (int j = 0; j < Y; ++j)
    {
        sum += layer1[j] * syn2[j * Z + l];
    }

    // Write back the computed value
    out[l] = 1.0f / (1.0f + exp(-sum));
}