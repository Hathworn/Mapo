#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Bprop2(const float* __restrict__ layer1, float* __restrict__ dsyn2, const float* __restrict__ out, const float alpha)
{
    int i = threadIdx.x; //256
    int j = blockDim.y * blockIdx.y + threadIdx.y; //10
    int k = blockIdx.x;  //Data.count

    // Utilize shared memory to minimize global memory access
    __shared__ float sharedLayer[256];

    // Load layer1 into shared memory
    sharedLayer[i] = layer1[256 * k + i];
    __syncthreads();

    // Perform computation and atomic addition
    atomicAdd(&dsyn2[i * 10 + j], out[k * 10 + j] * sharedLayer[i] * alpha);
}