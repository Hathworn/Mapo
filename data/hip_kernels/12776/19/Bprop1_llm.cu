#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Bprop1(const float* in, float* dsyn1, const float* dlayer1, const float alpha)
{
    int j = threadIdx.x;                         // 256
    int i = blockDim.y * blockIdx.y + threadIdx.y; // 28 * 28
    int k = blockIdx.x;                          // Data.count

    // Use shared memory to optimize memory access pattern.
    __shared__ float shared_dlayer1[256];
    if (j < 256) {
        shared_dlayer1[j] = dlayer1[k * 256 + j];
    }
    __syncthreads();

    float contribution = shared_dlayer1[j] * in[k * 28 * 28 + i] * alpha;

    // Reduce atomic contention by using private variable before atomicAdd.
    atomicAdd(&dsyn1[i * 256 + j], contribution);
}