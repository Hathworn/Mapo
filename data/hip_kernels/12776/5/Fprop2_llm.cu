#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Fprop2(const float* layer1, const float* syn2, float* out)
{
    int i = blockDim.y * blockIdx.y + threadIdx.y;  // Compute output neuron index
    int j = blockIdx.x;  // Data sample index

    float x = 0.0f;
    // Use shared memory for syn2 to improve access speed
    __shared__ float shared_syn2[256][10];

    int idx = threadIdx.x * 10 + i;  // Linear thread index for syn2
    if (i < 10) {
        // Load syn2 into shared memory
        for (int k = threadIdx.x; k < 256; k += blockDim.x) {
            shared_syn2[k][i] = syn2[k * 10 + i];
        }
    }
    __syncthreads();  // Ensure all syn2 data is loaded

    // Compute dot product using shared_syn2
    for (int k = 0; k < 256; ++k) {
        x += layer1[j * 256 + k] * shared_syn2[k][i];
    }

    out[j * 10 + i] = x;  // Write result to output
}