#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backProp1(float* in, float* dsyn1, float* layer1, float* syn2, float* label, float* out)
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int k = blockDim.y * blockIdx.y + threadIdx.y;
    float error = 0.0;

    // Use shared memory for performance improvement
    __shared__ float shared_label[10];
    __shared__ float shared_out[10];

    if (j < 10) {
        shared_label[j] = label[j];
        shared_out[j] = out[j];
    }
    __syncthreads();

    #pragma unroll
    for (int l = 0; l < 10; ++l) {
        error += (shared_label[l] - shared_out[l]) * syn2[k * 10 + l];
    }

    // Precompute repeated term for efficiency
    float term = layer1[k] * (1 - layer1[k]);
    float delta = error * term;
    atomicAdd(&dsyn1[j * 128 + k], delta * in[j] / 6000.0);
}