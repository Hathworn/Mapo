#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backProp2(float* layer1, float* dsyn2, float* label, float* out)
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int k = blockDim.y * blockIdx.y + threadIdx.y;

    // Check boundary conditions to ensure safe memory access
    if (j < 10 && k < 10) {
        // Precompute frequently used term
        float out_k = out[k];
        float delta = (label[k] - out_k) * (out_k * (1.0 - out_k));

        // Factor out constant division
        float updateValue = delta * layer1[j] * 0.00166667f; // (60000.0/10.0) = 6000, (1/6000) = 0.00166667

        atomicAdd(&dsyn2[j * 10 + k], updateValue); // Use atomicAdd to safely update dsyn2
    }
}