#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
    int32_t i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    if (i < N) {                                        // Ensure index is within bounds
        output[blockIdx.x * N + i] = K_A_B - row_sum[blockIdx.x] - col_sum[i];
    }
}