#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (; idx < num_elements; idx += stride) {
        // Unrolled memory access for efficiency
        to_normalize[idx * 3] = batch_index;
        to_normalize[idx * 3 + 1] = class_index;
        to_normalize[idx * 3 + 2] = static_cast<int64_t>(original[idx]);
    }
}