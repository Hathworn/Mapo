#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Precompute row offset to avoid redundant computation
    int32_t rowOffset = -row_sum[blockIdx.x];

    for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
        // Use direct indexing to improve readability and potential cache performance
        output[blockIdx.x * N + i] = rowOffset;
    }
}