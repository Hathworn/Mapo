#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Calculate global index and stride based on block and grid
    int32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride = blockDim.x * gridDim.x;

    // Use stride-based loop to effectively utilize all threads
    for (int32_t i = index; i < N; i += stride) {
        output[blockIdx.x * N + i] = -col_sum[i];
    }
}