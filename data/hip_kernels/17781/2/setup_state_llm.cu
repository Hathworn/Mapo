#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Utilize block and thread indices for unique state initialization
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread ID
    hiprand_init(seed, id, 0, &state[id]); // Assign each thread a unique state
}