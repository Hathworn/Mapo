#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Use blockIdx and blockDim to handle larger arrays
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) // Ensure within bounds
    {
        c[i] = a[i] + b[i];
    }
}