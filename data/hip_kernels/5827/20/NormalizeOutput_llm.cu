#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll the loop and use a while loop for better performance
    while (idx < num_elements) {
        to_normalize[idx * 3] = batch_index;
        to_normalize[idx * 3 + 1] = class_index;
        to_normalize[idx * 3 + 2] = static_cast<int64_t>(original[idx]);
        idx += blockDim.x * gridDim.x;
    }
}