#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Initialize the random state for each thread with a unique sequence offset
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique global thread index
    hiprand_init(seed, idx, 0, &state[idx]); // Use idx to ensure different states for each thread
}