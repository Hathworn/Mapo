#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gScatterAdd(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory for efficient access patterns
    __shared__ int sharedSparseIndices[BLOCK_DIM];
    __shared__ float sharedSparseData[BLOCK_DIM];

    if (idx < sparseSize) {
        // Load data into shared memory
        sharedSparseIndices[threadIdx.x] = sparseIndices[idx];
        sharedSparseData[threadIdx.x] = sparseData[idx];
        __syncthreads();

        // Check bounds and perform addition
        if (sharedSparseIndices[threadIdx.x] >= -offset && sharedSparseIndices[threadIdx.x] + offset < denseSize) {
            atomicAdd(&denseData[sharedSparseIndices[threadIdx.x] + offset], sharedSparseData[threadIdx.x]);
        }
    }
}