#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gCopyCols(float* out, const float* in, size_t rows, size_t colsIn, const size_t* sourceColIdx, size_t colsOut) {
    int j = blockIdx.x + blockIdx.y * gridDim.x; // Better 2D grid launch for large rows
    if (j < rows) {
        const float* rowIn = in + j * colsIn;
        float* rowOut = out + j * colsOut;

        for (int tid = threadIdx.x; tid < colsOut; tid += blockDim.x) { // Start thread loop with threadIdx
            rowOut[tid] = rowIn[sourceColIdx[tid]];
        }
    }
}