#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline float stableSigmoid(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gLSTMOutputBackward(float* outCell, float* outXW, float* outSU, float* outB, 
                                    const float* cell, const float* xW, const float* sU, 
                                    const float* b, const float* adj, size_t rows, size_t cols) {
    // Optimize grid-stride loop
    for(int j = blockIdx.x; j < rows; j += gridDim.x) {
        float* rowOutCell = outCell + j * cols;
        float* rowOutXW = outXW + j * cols * 4;
        float* rowOutSU = outSU + j * cols * 4;

        const float* rowCell = cell + j * cols;
        const float* xWrow = xW + j * cols * 4;
        const float* sUrow = sU + j * cols * 4;
        const float* rowAdj = adj + j * cols;

        // Optimize grid-stride loop
        for(int i = threadIdx.x; i < cols; i += blockDim.x) {
            int k = i + 3 * cols;
            float go = stableSigmoid(xWrow[k] + sUrow[k] + b[k]);

            float t = tanhf(rowCell[i]);
            float adj = rowAdj[i];

            // Simplified redundant multiplication
            float commonTerm = t * go * (1 - go) * adj;

            // dc/dc_{t-1}
            if(outCell)
                rowOutCell[i] += go * (1 - t * t) * adj;

            // dc/d(b_o) = dc/d(xW_f) ...
            if(outXW)
                rowOutXW[k] += commonTerm;
            if(outSU)
                rowOutSU[k] += commonTerm;
            if(outB)
                atomicAdd(outB + k, commonTerm);
        }
    }
}