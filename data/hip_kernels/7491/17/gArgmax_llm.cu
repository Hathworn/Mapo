#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gArgmax(float* out, const float* data, size_t rows, size_t cols) {
    size_t row = blockIdx.x;
    size_t startInd = row * cols;
    float maxScore = -99999;
    size_t maxInd;

    // Use shared memory to improve memory access efficiency
    extern __shared__ float sharedData[];
    size_t tid = threadIdx.x;

    if (tid < cols) {
        sharedData[tid] = data[startInd + tid];
    }
    __syncthreads();

    for (size_t col = 0; col < cols; ++col) {
        float score = sharedData[col];
        if (score > maxScore) {
            maxScore = score;
            maxInd = col;
        }
    }

    out[row] = maxInd;
}