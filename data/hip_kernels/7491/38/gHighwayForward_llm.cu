#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableSigmoid(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gHighwayForward(float* out, const float* in1, const float* in2, const float* t, size_t length) {
    // Use a single loop iteration and calculate the index directly
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    for(int offset = index; offset < length; offset += blockDim.x * gridDim.x) {
        float sigma = stableSigmoid(t[offset]);
        out[offset] = in1[offset] * sigma + in2[offset] * (1.f - sigma);
    }
}