#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gPasteCols(float* out, const float* in, size_t rows, size_t colsOut, const size_t* targetColIdx, size_t colsIn) {
    // Utilize grid-stride loop for better parallelism over rows
    for(int j = blockIdx.x + blockIdx.y * gridDim.x; j < rows; j += gridDim.x * gridDim.y) {
        const float* rowIn = in + j * colsIn;
        float* rowOut = out + j * colsOut;

        // Utilize grid-stride loop for column processing
        for(int i = threadIdx.x; i < colsIn; i += blockDim.x) {
            if(i < colsIn) {
                rowOut[targetColIdx[i]] += rowIn[i];
            }
        }
    }
}