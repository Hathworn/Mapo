#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gCopyRows(float* out, const float* in, size_t cols, const size_t* sourceRowIdx, size_t rows) {
    int j = blockIdx.x + blockIdx.y * gridDim.x; // Calculate row index using 2D grid
    while (j < rows) {
        size_t dstId = j;
        size_t srcId = sourceRowIdx[j];
        float* rowOut = out + dstId * cols;
        const float* rowIn = in + srcId * cols;
        
        for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) { // Improve parallel efficiency by starting directly with threadIdx.x
            rowOut[tid] = rowIn[tid];
        }
        
        j += gridDim.x * gridDim.y; // Increment row index by 2D grid size
    }
}