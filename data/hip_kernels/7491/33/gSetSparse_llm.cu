#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gSetSparse(float* out, const size_t* indices, const float* values, int length) {
    // Calculate the global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a single loop, thread processes its own data
    if(index < length) {
        out[indices[index]] = values[index];
    }
}