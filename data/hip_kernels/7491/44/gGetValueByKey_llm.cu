#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gGetValueByKey(float* d_in, float* d_out, int* indeces, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index

    if (tid < n) {
        int index = indeces[tid]; // Get index for this thread
        d_out[tid] = d_in[index]; // Write the corresponding d_in value to d_out
    }
}