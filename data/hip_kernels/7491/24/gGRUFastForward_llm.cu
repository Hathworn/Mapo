#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableSigmoid(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gGRUFastForward(float* out, const float* state, const float* xW, const float* sU, const float* b, const float* mask, size_t rows, size_t cols, bool final) {
    int j = blockIdx.x;
    if(j < rows) {
        float m = !mask || mask[j];
        float* rowOut = out + j * cols;
        const float* rowState = state + j * cols;
        const float* xWrow = xW + j * cols * 3;
        const float* sUrow = sU + j * cols * 3;

        for(int i = threadIdx.x; i < cols; i += blockDim.x) {
            // Precompute reused indices for readability
            int k = i + cols;
            int l = i + 2 * cols;

            // Compute the r and z gates using sigmoid function
            float r = stableSigmoid(xWrow[i] + sUrow[i] + b[i]);
            float z = stableSigmoid(xWrow[k] + sUrow[k] + b[k]);

            // Compute the candidate hidden state h_t
            float h;
            if(final)
                h = tanhf(xWrow[l] + (sUrow[l] + b[l]) * r);
            else
                h = tanhf(xWrow[l] + sUrow[l] * r + b[l]);

            // Compute the final output for this element
            float outVal = (1.0f - z) * h + z * rowState[i];
            rowOut[i] = m * outVal + (1 - m) * rowState[i];
        }
    }
}