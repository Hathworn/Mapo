#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gLogSoftmaxGrad(float* grad, const float* adj, const float* val, const int rows, const int cols) {
    const int j = blockIdx.x + blockIdx.y * gridDim.x;
    if (j < rows) {
        extern __shared__ float _shared[];
        float* _sum = _shared;

        float* gradRow = grad + j * cols;
        const float* adjRow = adj + j * cols;
        const float* valRow = val + j * cols;

        // Initialize shared memory for sum
        float thread_sum = 0.0;
        for(int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
            thread_sum += adjRow[tid];
        }
        _sum[threadIdx.x] = thread_sum;
        __syncthreads();

        // Reduce sum across the block
        for(int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
            if(threadIdx.x < stride) {
                _sum[threadIdx.x] += _sum[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Broadcast the computed sum to all threads
        float sum_val = _sum[0];
        for(int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
            gradRow[tid] += adjRow[tid] - (expf(valRow[tid]) * sum_val);
        }
    }
}