#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gSoftmaxGrad(float* grad, const float* adj, const float* val, const int rows, const int cols) {
  // Calculate row index using block and grid dimensions
  int j = blockIdx.x + blockIdx.x + gridDim.x;
  if (j < rows) {
    extern __shared__ float _share[];
    float* _sum = _share;
    
    float* gradRow = grad + j * cols;
    const float* adjRow = adj + j * cols;
    const float* valRow = val + j * cols;
    
    // Use local variable to accumulate sum
    float localSum = 0.0f;
    for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
      localSum += valRow[tid] * adjRow[tid];
    }
    
    _sum[threadIdx.x] = localSum;
    __syncthreads();
    
    // Use reduction to calculate sum
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
      if (threadIdx.x < stride) {
        _sum[threadIdx.x] += _sum[threadIdx.x + stride];
      }
      __syncthreads();
    }
    
    float totalSum = _sum[0];
    
    // Calculate grad values
    for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) {
      float val = valRow[tid] * (adjRow[tid] - totalSum);
      if (val != 0.0f) {
        gradRow[tid] += val;
      }
    }
  }
}