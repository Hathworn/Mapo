#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableSigmoid(float x) {
    if (x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gLSTMCellForward(float* out, const float* cell, const float* xW, const float* sU, const float* b, const float* mask, size_t rows, size_t cols) {
    // Calculate global row index
    int j = blockIdx.x + blockIdx.y * gridDim.x; // Use 2D grid for better distribution

    if (j < rows) {
        float m = !mask || mask[j];
        float* rowOut = out + j * cols;
        const float* rowCell = cell + j * cols;
        const float* xWrow = xW + j * cols * 4;
        const float* sUrow = sU + j * cols * 4;

        for (int i = threadIdx.x; i < cols; i += blockDim.x) { // Loop unrolling potential for better efficiency
            float gf = stableSigmoid(xWrow[i] + sUrow[i] + b[i]);
        
            int k = i + cols;
            float gi = stableSigmoid(xWrow[k] + sUrow[k] + b[k]);
        
            int l = i + 2 * cols;
            float gc = tanhf(xWrow[l] + sUrow[l] + b[l]);
        
            float cout = gf * rowCell[i] + gi * gc;
            rowOut[i] = m * cout + (1 - m) * rowCell[i];
        }
    }
}