#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gMaxPoolingForward(float* out, int outRows, int outCols, float* in, int inRows, int inCols, float* mask, int numKernels, int maskCols, int width, int lastWidth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    if(tid >= outRows * outCols)
        return;

    int rowId = tid / outCols;  // Correct calculation for rowId using outCols
    int colId = tid % outCols;  // Correct calculation for colId using outCols

    // Calculate starting position in the input matrix and mask
    float* b = in + (rowId * inCols) + (colId * width);
    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;

    // Adjust width for last row
    if(colId == outCols - 1) {
        width = lastWidth;
    }

    float currentMax = b[0] * localMask[0];
    for(int i = 1; i < width; ++i) {
        float product = b[i] * localMask[i];
        if(product > currentMax) {
            currentMax = product;
        }
    }

    out[rowId * outCols + colId] = currentMax;  // Correct output index calculation
}