#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void gGather(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < sparseSize) {  // Consolidated boundary check
        int index = sparseIndices[idx] + offset;  // Precompute combined index
        if (index >= 0 && index < denseSize) {  // Use precomputed index and simplified condition
            sparseData[idx] = denseData[index];
        }
    }
}