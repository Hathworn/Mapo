#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableSigmoid(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gLSTMCellBackward(float* outCell, float* outXW, float* outSU, float* outB, const float* cell, const float* xW, const float* sU, const float* b, const float* mask, const float* adj, size_t rows, size_t cols) {
    // Pre-compute indices to improve warp efficiency
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stepRow = gridDim.x * blockDim.x;
    int stepCol = blockDim.x;

    for(int j = tid; j < rows; j += stepRow) {
        float m = !mask || mask[j]; // Mask condition computed outside loop

        float* rowOutCell = outCell + j * cols;
        float* rowOutXW = outXW + j * cols * 4;
        float* rowOutSU = outSU + j * cols * 4;

        const float* rowCell = cell + j * cols;
        const float* xWrow = xW + j * cols * 4;
        const float* sUrow = sU + j * cols * 4;
        const float* rowAdj = adj + j * cols;

        for(int i = threadIdx.x; i < cols; i += stepCol) {
            float gf = stableSigmoid(xWrow[i] + sUrow[i] + b[i]);
            int k = i + cols;
            float gi = stableSigmoid(xWrow[k] + sUrow[k] + b[k]);
            int l = i + 2 * cols;
            float gc = tanhf(xWrow[l] + sUrow[l] + b[l]);
            float adj = rowAdj[i];

            // Optimized condition checks by combining and minimizing out-of-bound checks
            if(outCell) rowOutCell[i] += (m * gf - m + 1) * adj;
            if(m) { // Factor out m to minimize conditional checks
                float dcdxf = rowCell[i] * gf * (1 - gf) * adj;
                float dcdb_i = gc * gi * (1 - gi) * adj;
                float dcdxc = gi * (1 - gc * gc) * adj;

                if(outXW) {
                    rowOutXW[i] += dcdxf;
                    rowOutXW[k] += dcdb_i;
                    rowOutXW[l] += dcdxc;
                }
                if(outSU) {
                    rowOutSU[i] += dcdxf;
                    rowOutSU[k] += dcdb_i;
                    rowOutSU[l] += dcdxc;
                }
                if(outB) {
                    atomicAdd(outB + i, dcdxf);
                    atomicAdd(outB + k, dcdb_i);
                    atomicAdd(outB + l, dcdxc);
                }
            }
        }
    }
}