#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gScatterUpdate(float* denseData, float* sparseData, int* sparseIndices, int denseSize, int sparseSize, int offset) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (idx < sparseSize) {
        // Calculate the position to update in denseData
        int denseIndex = sparseIndices[idx] + offset;
        // Ensure the calculated index is within valid range
        if (denseIndex >= 0 && denseIndex < denseSize) {
              denseData[denseIndex] = sparseData[idx];
        }
    }
}