#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gMaxPoolingBackward(float* adj, int adjRows, int adjCols, float* in, 
                                    float* adjIn, int inRows, int inCols, float* mask, 
                                    int numKernels, int maskCols, int width, int lastWidth) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Early exit for threads outside the valid range
    if (tid >= adjRows * adjCols) return;

    int rowId = tid / adjCols; // Corrected computation of rowId
    int colId = tid % adjCols; // Corrected computation of colId

    int currWidth = (colId == adjCols - 1) ? lastWidth : width; // Adjusted criteria using adjCols

    float* b = in + (rowId * inCols) + (colId * width);
    float* localMask = mask + (rowId / numKernels) * maskCols + colId * width;

    size_t currentMaxIdx = 0;

    // Loop unrolling for small fixed widths might improve performance
    for (int i = 1; i < currWidth; ++i) {
        if (b[i] * localMask[i] > b[currentMaxIdx] * localMask[currentMaxIdx]) {
            currentMaxIdx = i;
        }
    }

    // Reduce accesses to global memory
    atomicAdd(&adjIn[(rowId * inCols) + (colId * width) + currentMaxIdx], 
              adj[rowId + (colId * adjCols)]);
}