#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gPasteRows(float* out, const float* in, size_t cols, const size_t* targetRowIdx, size_t rows) {
    // Optimized iteration for blocks
    int j = blockIdx.x + gridDim.x * blockIdx.y;
    while (j < rows) {
        size_t dstId = targetRowIdx[j];
        size_t srcId = j;
        
        float* rowOut = out + dstId * cols;
        const float* rowIn = in + srcId * cols;

        // Optimized iteration for threads
        int i = threadIdx.x;
        while (i < cols) {
            atomicAdd(rowOut + i, rowIn[i]);
            i += blockDim.x;
        }
        
        j += gridDim.x * gridDim.y;
    }
}