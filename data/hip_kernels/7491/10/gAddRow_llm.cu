#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gAddRow(float* out, const float* in, int length) {
    // Calculate global index
    int index = blockDim.x * blockIdx.x + threadIdx.x;

    // Loop with step size of total threads
    for (int i = index; i < length; i += blockDim.x * gridDim.x) {
        out[i] = in[i] + out[i];
    }
}