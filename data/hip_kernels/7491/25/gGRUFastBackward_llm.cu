#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableSigmoid(float x) {
  if (x >= 0) {
    float z = expf(-x);
    return 1.0 / (1.0 + z);
  } else {
    float z = expf(x);
    return z / (1.0 + z);
  }
}

__global__ void gGRUFastBackward(float* outState, float* outXW, float* outSU, float* outB, const float* state, const float* xW, const float* sU, const float* b, const float* mask, const float* adj, size_t rows, size_t cols, bool final) {
  for (int bid = blockIdx.x; bid < rows; bid += gridDim.x) { // Optimize grid-stride loop
    float m = !mask || mask[bid];
    
    float* rowOutState = outState + bid * cols;
    float* rowOutXW = outXW + bid * cols * 3;
    float* rowOutSU = outSU + bid * cols * 3;

    const float* rowState = state + bid * cols;
    const float* rowXW = xW + bid * cols * 3;
    const float* rowSU = sU + bid * cols * 3;
    const float* rowAdj = adj + bid * cols;

    for (int tid = threadIdx.x; tid < cols; tid += blockDim.x) { // Optimize block-stride loop
      int i = tid;
      int k = i + cols;
      int l = i + 2 * cols;

      float r = stableSigmoid(rowXW[i] + rowSU[i] + b[i]);
      float z = stableSigmoid(rowXW[k] + rowSU[k] + b[k]);

      float h;
      if (final)
        h = tanhf(rowXW[l] + (rowSU[l] + b[l]) * r);
      else
        h = tanhf(rowXW[l] + rowSU[l] * r + b[l]);

      float adj = rowAdj[i];
      float t = (1 - z) * (1 - h * h);

      // df/ds
      if (outState)
        rowOutState[i] += (m * z - m + 1) * adj;

      // df/d(xW_r) ...
      float dfdxW_r = m * r * (1 - r) * t * adj;
      dfdxW_r *= final ? (rowSU[l] + b[l]) : rowSU[l];
      if (outXW)
        rowOutXW[i] += dfdxW_r;
      if (outSU)
        rowOutSU[i] += dfdxW_r;
      if (outB)
        atomicAdd(outB + i, dfdxW_r);

      // df/d(xW_z) ...
      float dfdxW_z = m * (1 - z) * z * (rowState[i] - h) * adj;
      if (outXW)
        rowOutXW[k] += dfdxW_z;
      if (outSU)
        rowOutSU[k] += dfdxW_z;
      if (outB)
        atomicAdd(outB + k, dfdxW_z);

      // df/d(xW_x) ...
      float dfdxW_x = m * t * adj;
      if (outXW)
        rowOutXW[l] += dfdxW_x;
      if (outSU)
        rowOutSU[l] += dfdxW_x * r;
      if (outB) {
        if (final)
          atomicAdd(outB + l, dfdxW_x * r);
        else
          atomicAdd(outB + l, dfdxW_x);
      }
    }
  }
}