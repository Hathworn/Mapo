#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline float stableSigmoid(float x) {
    if(x >= 0) {
        float z = expf(-x);
        return 1.0 / (1.0 + z);
    } else {
        float z = expf(x);
        return z / (1.0 + z);
    }
}

__global__ void gLSTMOutputForward(float* out, const float* cell, const float* xW, const float* sU, const float* b, size_t rows, size_t cols) {
    // Calculate global thread index
    int row = blockIdx.x + blockIdx.y * gridDim.x;
    int col = threadIdx.x;

    if(row < rows && col < cols) {
        int baseIndex = row * cols;
        float* rowOut = out + baseIndex;
        const float* rowCell = cell + baseIndex;

        int xWIndex = baseIndex * 4 + col + 3 * cols;
        const float* xWrow = xW + xWIndex;
        const float* sUrow = sU + xWIndex;

        float go = stableSigmoid(xWrow[0] + sUrow[0] + b[col + 3 * cols]);
        rowOut[col] = go * tanhf(rowCell[col]);
    }
}