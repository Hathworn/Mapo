#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void expandPlanes_kernel_Fp32_NCHW(float* output, const uint64_t* masks, const float* values, int n) {
    // Block size of 256, same mask/val for 64 consecutive threads.
    constexpr int kNumShmemElements = 256 / 64;

    __shared__ uint64_t shMasks[kNumShmemElements];
    __shared__ float shVals[kNumShmemElements];

    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int planeIndex = index >> 6;

    if (planeIndex >= n) return;

    // Load inputs to shared memory only if necessary.
    // Optimize memory access patterns.
    int shIndex = threadIdx.x >> 6;
    if (threadIdx.x < kNumShmemElements * 64) {
        if (threadIdx.x < kNumShmemElements) {
            shMasks[threadIdx.x] = masks[planeIndex + threadIdx.x];
            shVals[threadIdx.x] = values[planeIndex + threadIdx.x];
        }
        __syncthreads();

        uint64_t mask = shMasks[shIndex];
        int sqIndex = index & 0x3F;
        
        // Utilize registers for temporary variables
        float op = (mask & (1ull << sqIndex)) ? shVals[shIndex] : 0.0f;
        output[index] = op;
    }
}