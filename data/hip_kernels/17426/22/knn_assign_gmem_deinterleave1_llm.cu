#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void knn_assign_gmem_deinterleave1(uint32_t length, uint16_t k, uint32_t *neighbors) {
    uint64_t sample = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early exit for out-of-bounds threads
    if (sample >= length) return;
    
    // Calculate base indices once
    uint64_t baseIndex = sample * k;
    uint64_t sourceIndex = sample * 2 * k;
    
    // Use shared memory for better performance if appropriate (limited by SM size)
    __shared__ uint32_t sharedNeighbors[1024]; // Example size

    if (sample % 2 == 1) {
        for (int i = threadIdx.x; i < k; i += blockDim.x) { // Work division among threads in a block
            neighbors[baseIndex + i] = neighbors[sourceIndex + i];
        }
    } else {
        uint64_t targetIndex = (length + sample) * k + k;
        for (int i = threadIdx.x; i < k; i += blockDim.x) { // Work division among threads in a block
            neighbors[targetIndex + i] = neighbors[sourceIndex + i];
        }
    }
}