#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void knn_assign_gmem_deinterleave2( uint32_t length, uint16_t k, uint32_t *neighbors) {
    // Calculate initial sample index
    uint64_t sample = blockIdx.x * blockDim.x + threadIdx.x;
    sample *= 2;
    
    // Early exit if sample is out of range
    if (sample >= length) return;

    // Use a pointer offset to improve memory access pattern
    uint32_t *source = &neighbors[(length + sample) * k + k];
    uint32_t *dest = &neighbors[sample * k];

    // Unroll loop to reduce loop overhead
    #pragma unroll
    for (int i = 0; i < k; i++) {
        dest[i] = source[i];
    }
}