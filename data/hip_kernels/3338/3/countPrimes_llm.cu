#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define number_type unsigned long long

const int block_size = 1024; // 2**10 threads
const int thread_size = 32768 * 2 * 2; // 2**15 max elements per thread always keep even number
const number_type max_chunk_size = pow(2, 31) + pow(2, 30); // 2**31 items cause reduce ram use else failed allocations, always keep even number

hipError_t find_primes_cuda(number_type n, number_type r);

void set_one(char* dev_arr, unsigned int size);
template <typename T>
void reset(T* dev_arr, size_t count);

template <typename T>
T* device(size_t count);
template <typename T>
T* host(size_t count);
void confirmCudaNoError();
void cudaWait();
template <typename T>
T* to_host(const T* dev_ptr, size_t count, T* host_ptr = nullptr);
template <typename T>
T* to_device(const T* host_ptr, size_t count, T* dev_ptr = nullptr);

__global__ void countPrimes(char* dev_chunk, number_type* count_accumulation_chunk, const number_type startValue, const number_type endValue, const int thread_size)
{
    const auto my_thread_id = blockIdx.x * block_size + threadIdx.x;
    auto my_start_value = startValue + my_thread_id * thread_size;
    auto my_end_value = my_start_value + thread_size;
    
    // Ensure the end value does not exceed the overall end value
    if (my_end_value > endValue) {
        my_end_value = endValue;
    }
    
    unsigned long count = 0;
    
    // Adjust to account for the first prime number
    if (my_start_value == 0) {
        count += 1; // Account for prime number 2
        my_start_value += 1; // Increment to skip immediate next even number
    }
    
    // Ensure starting with an odd number
    if (my_start_value % 2 == 0) {
        my_start_value += 1;
    }
    
    // Iterate over odd numbers only to count primes
    for (auto i = my_start_value; i < my_end_value; i += 2) {
        const auto current_status = dev_chunk[i - startValue];
        if (current_status == 1) {
            count += 1;
        }
    }
    
    // Store result in corresponding thread index
    count_accumulation_chunk[my_thread_id] = count;
}