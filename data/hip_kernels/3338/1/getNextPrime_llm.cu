#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define number_type unsigned long long

const int block_size = 1024; // 2**10 threads
const int thread_size = 32768 * 2 * 2; // 2**15 max elements per thread always keep even number
const number_type max_chunk_size = pow(2, 31) + pow(2, 30); // 2**31 items cause reduce ram use else failed allocations, always keep even number

hipError_t find_primes_cuda(number_type n, number_type r);

void set_one(char* dev_arr, unsigned int size);
template <typename T>
void reset(T* dev_arr, size_t count);

template <typename T>
T* device(size_t count);
template <typename T>
T* host(size_t count);
void confirmCudaNoError();
void cudaWait();
template <typename T>
T* to_host(const T* dev_ptr, size_t count, T* host_ptr = nullptr);
template <typename T>
T* to_device(const T* host_ptr, size_t count, T* dev_ptr = nullptr);

__global__ void getNextPrime(number_type* dev_temp_min_primes, int size, number_type* d_ans)
{
    // Use shared memory to reduce global memory access
    __shared__ number_type shared_mem[block_size];
    int threadId = threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Load elements into shared memory
    for (int i = threadId; i < size; i += stride) {
        shared_mem[threadId] = dev_temp_min_primes[i];
        __syncthreads();

        // Find the first non-zero element
        if (threadId == 0) {
            for (int j = 0; j < blockDim.x; ++j) {
                if (shared_mem[j] != 0) {
                    *d_ans = shared_mem[j];
                    return;
                }
            }
        }
    }
}