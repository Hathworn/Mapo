#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define number_type unsigned long long

const int block_size = 1024; // 2**10 threads
const int thread_size = 32768 * 2 * 2; // 2**15 max elements per thread always keep even number
const number_type max_chunk_size = pow(2, 31) + pow(2, 30); // 2**31 items cause reduce ram use else failed allocations, always keep even number

hipError_t find_primes_cuda(number_type n, number_type r);

void set_one(char* dev_arr, unsigned int size);
template <typename T>
void reset(T* dev_arr, size_t count);

template <typename T>
T* device(size_t count);
template <typename T>
T* host(size_t count);
void confirmCudaNoError();
void cudaWait();
template <typename T>
T* to_host(const T* dev_ptr, size_t count, T* host_ptr = nullptr);
template <typename T>
T* to_device(const T* host_ptr, size_t count, T* dev_ptr = nullptr);

__global__ void getNextPrimeFast(char* dev_chunk, number_type currentValue, const number_type startValue, const number_type endValue, number_type* d_ans)
{
    // Using shared memory to reduce global memory accesses
    __shared__ bool found;
    found = false;
    __syncthreads();

    // Distribute work among threads
    for (auto i = startValue + threadIdx.x + blockIdx.x * blockDim.x; i < endValue; i += blockDim.x * gridDim.x)
    {
        // Check only if not found by any thread
        if (!found && dev_chunk[i] == 1)
        {
            *d_ans = i;
            found = true; // Update shared variable
        }
        __syncthreads(); // Ensure all threads see the update
        if (found) return; // Early exit if prime found
    }
}