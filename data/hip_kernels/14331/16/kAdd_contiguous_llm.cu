#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Use shared memory to improve cache efficiency
    __shared__ float shared_a[256];
    __shared__ float shared_b[256];

    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < numEls; i += numThreads) {
        // Load elements into shared memory
        shared_a[threadIdx.x] = a[i];
        shared_b[threadIdx.x] = b[i];
        __syncthreads();

        // Perform addition
        dest[i] = shared_a[threadIdx.x] + shared_b[threadIdx.x];
        __syncthreads();
    }
}