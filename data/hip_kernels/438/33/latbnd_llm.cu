#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sign(DECNUM x) {
    return ((x > 0.0f) - (x < 0.0f));
}

__device__ int mminus2(int ix, int nx) {
    return (ix <= 1) ? 0 : ix - 2;
}

__device__ int pplus(int ix, int nx) {
    return (ix == nx - 1) ? nx - 1 : ix + 1;
}

__device__ int mminus(int ix, int nx) {
    return (ix == 0) ? 0 : ix - 1;
}

__global__ void latbnd(int nx, int ny, DECNUM * uu) {
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int i = ix + iy * nx;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ DECNUM uut[16][16];
    __shared__ DECNUM uub[16][16];

    if (ix < nx && iy < ny) {
        unsigned int yplus = pplus(iy, ny);

        // Load boundary values only on first and last rows
        uut[tx][ty] = uu[ix + yplus * nx];
        if (iy == 0) {
            uu[i] = uut[tx][ty];
        }

        if (iy == ny - 1) {
            unsigned int yminus = mminus(iy, ny);
            uub[tx][ty] = uu[ix + yminus * nx];
            uu[i] = uub[tx][ty];
        }
    }
}