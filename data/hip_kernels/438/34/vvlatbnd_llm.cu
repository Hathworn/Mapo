#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int sign(DECNUM x) {
    return ((x > 0.0f) - (x < 0.0f));
}

__device__ int mminus2(int ix, int nx) {
    return (ix <= 1) ? 0 : ix - 2; // Simplified conditional return
}

__device__ int pplus(int ix, int nx) {
    return (ix == nx - 1) ? nx - 1 : ix + 1; // Simplified conditional return
}

__device__ int mminus(int ix, int nx) {
    return (ix == 0) ? 0 : ix - 1; // Simplified conditional return
}

__global__ void vvlatbnd(int nx, int ny, DECNUM *uu) {
    // Neumann_v boundary
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int i = ix + iy * nx;

    __shared__ DECNUM uut[16][16];
    __shared__ DECNUM uub[16][16];

    if (ix < nx && iy < ny) {
        unsigned int yminus = mminus(iy, ny);
        unsigned int yplus = pplus(iy, ny);

        // Use fewer shared memory accesses and simplified boundary logic
        uut[threadIdx.x][threadIdx.y] = uu[ix + yplus * nx];
        uub[threadIdx.x][threadIdx.y] = uu[ix + yminus * nx];

        if (iy == 0) {
            uu[i] = uut[threadIdx.x][threadIdx.y];
        } else if (iy == ny - 2) {
            uu[i] = uub[threadIdx.x][threadIdx.y];
        }
    }
}