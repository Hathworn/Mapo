#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void golGpu(int height, int width, unsigned char* pBuffer1, unsigned char* pBuffer2) {
    // Calculate global thread x and y position
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within the bounds
    if(x < height && y < width) { 
        int indx = x * width + y;

        pBuffer2[indx] = pBuffer1[indx];

        int num = 0;
        
        // Loop to calculate live neighbors
        for(int dx = -1; dx <= 1; ++dx) {
            for(int dy = -1; dy <= 1; ++dy) {
                if(dx != 0 || dy != 0) { // Exclude the cell itself
                    int nx = x + dx;
                    int ny = y + dy;
                    if(nx >= 0 && nx < height && ny >= 0 && ny < width) {
                        num += pBuffer1[nx * width + ny];
                    }
                }
            }
        }

        // Apply the rules of the Game of Life
        if(num < 2 || num > 3) 
            pBuffer2[indx] = 0x0;
        else if(num == 3) 
            pBuffer2[indx] = 0x1;
    }
}