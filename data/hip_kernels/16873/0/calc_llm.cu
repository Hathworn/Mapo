#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define GravConst 6.674e-11
#define EPS 1e-6
#define POINTS_SIZE 10

__device__ inline float sqr(float x) { return x * x; }

__global__ void calc(float* points, float* resPoints, int dt, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tix = threadIdx.x;
    int steps = (size + 127) / 128;  // Calculate number of steps directly

    float fx, fy, fz;
    float x, y, z, m, vx, vy, vz, ax, ay, az;

    if (idx < size) {
        x = points[idx * POINTS_SIZE + 0];
        y = points[idx * POINTS_SIZE + 1];
        z = points[idx * POINTS_SIZE + 2];
        m = points[idx * POINTS_SIZE + 3];
        vx = points[idx * POINTS_SIZE + 4];
        vy = points[idx * POINTS_SIZE + 5];
        vz = points[idx * POINTS_SIZE + 6];
        ax = points[idx * POINTS_SIZE + 7];
        ay = points[idx * POINTS_SIZE + 8];
        az = points[idx * POINTS_SIZE + 9];

        for (int iteration = 0; iteration < steps; iteration++) {
            __shared__ float cached_points[128 * POINTS_SIZE];

            // Load data into shared memory with stride
            if (iteration * 128 + tix < size) {
                for (int i = 0; i < POINTS_SIZE; i++) {
                    cached_points[tix * POINTS_SIZE + i] = points[(iteration * 128 + tix) * POINTS_SIZE + i];
                }
            }
            __syncthreads();

            fx = fy = fz = 0;
            for (int i = 0; i < 128; i++) {
                int currentIdx = iteration * 128 + i;
                if (currentIdx < size && currentIdx != idx) {
                    float dx = cached_points[i * POINTS_SIZE] - x;
                    float dy = cached_points[i * POINTS_SIZE + 1] - y;
                    float dz = cached_points[i * POINTS_SIZE + 2] - z;
                    float dist = sqrt(dx * dx + dy * dy + dz * dz) + EPS;  // Added EPS directly
                    float F = (GravConst * m * cached_points[i * POINTS_SIZE + 3]) / (dist * dist);
                    fx += F * dx / dist;
                    fy += F * dy / dist;
                    fz += F * dz / dist;
                }
            }
            __syncthreads();  // Ensure all threads complete computation before global write

            resPoints[idx * POINTS_SIZE] = x + vx * dt + (ax * sqr(dt)) / 2;
            resPoints[idx * POINTS_SIZE + 1] = y + vy * dt + (ay * sqr(dt)) / 2;
            resPoints[idx * POINTS_SIZE + 2] = z + vz * dt + (az * sqr(dt)) / 2;
            resPoints[idx * POINTS_SIZE + 3] = m;
            resPoints[idx * POINTS_SIZE + 4] = vx + ax * dt;
            resPoints[idx * POINTS_SIZE + 5] = vy + ay * dt;
            resPoints[idx * POINTS_SIZE + 6] = vz + az * dt;
            resPoints[idx * POINTS_SIZE + 7] = fx / m;
            resPoints[idx * POINTS_SIZE + 8] = fy / m;
            resPoints[idx * POINTS_SIZE + 9] = fz / m;
        }
    }
}