#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_Comput (int *h, int N, int T) {
    // Calculate linear global thread ID
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int index = row * N + col;  // Efficient index calculation

    // Loop through T iterations
    for (int t = 0; t < T; t++) {
        h[index] = index;  // Store the computed index directly
    }
}