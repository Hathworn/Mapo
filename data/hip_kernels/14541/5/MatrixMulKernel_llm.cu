#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
{
    // Calculate the row index of the Pd element and M
    int Row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    // Calculate the column index of Pd and N
    int Col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    float Pvalue = 0;
    // Use shared memory to optimize memory access
    __shared__ float Mds[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Nds[BLOCK_SIZE][BLOCK_SIZE];
  
    for (int tileIdx = 0; tileIdx < Width / BLOCK_SIZE; ++tileIdx) {
        // Load data into shared memory from Md and Nd
        Mds[threadIdx.y][threadIdx.x] = Md[Row * Width + (tileIdx * BLOCK_SIZE + threadIdx.x)];
        Nds[threadIdx.y][threadIdx.x] = Nd[(tileIdx * BLOCK_SIZE + threadIdx.y) * Width + Col];
        __syncthreads(); // Synchronize threads to ensure all data is loaded

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            // Accumulate results for a block
            Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        }
        __syncthreads(); // Synchronize threads before loading new data
    }

    Pd[Row * Width + Col] = Pvalue;
}