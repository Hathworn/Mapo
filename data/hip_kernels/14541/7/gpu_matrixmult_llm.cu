#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_matrixmult(int *gpu_a, int *gpu_b, int *gpu_c, int N) {
    // Calculate global row and column indices
    int col = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;

    // Ensure within matrix bounds
    if(col < N && row < N) {
        int sum = 0; // Initialize sum to zero

        // Use shared memory for tile storage
        __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

        for(int m = 0; m < (N + BLOCK_SIZE - 1) / BLOCK_SIZE; ++m) {
            // Load tiles into shared memory
            if (row < N && m * BLOCK_SIZE + threadIdx.x < N) {
                tile_a[threadIdx.y][threadIdx.x] = gpu_a[row * N + m * BLOCK_SIZE + threadIdx.x];
            } else {
                tile_a[threadIdx.y][threadIdx.x] = 0;
            }

            if (col < N && m * BLOCK_SIZE + threadIdx.y < N) {
                tile_b[threadIdx.y][threadIdx.x] = gpu_b[(m * BLOCK_SIZE + threadIdx.y) * N + col];
            } else {
                tile_b[threadIdx.y][threadIdx.x] = 0;
            }

            __syncthreads();

            // Compute partial product
            for(int k = 0; k < BLOCK_SIZE; ++k) {
                sum += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
            }

            __syncthreads();
        }

        // Write the result to gpu_c
        gpu_c[row * N + col] = sum;
    }
}
```
