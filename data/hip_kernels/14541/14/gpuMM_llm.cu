#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpuMM(double *A, double *B, double *C, int N)
{
    // Matrix multiplication for NxN matrices C=A*B
    // Each thread computes a single element of C
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) // Check to ensure row and col are within bounds
    {
        double sum = 0.0; // Use double to match input data type
        for (int n = 0; n < N; ++n)
        {
            sum += A[row * N + n] * B[n * N + col];
        }
        C[row * N + col] = sum;
    }
}