#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpuMM(float *A, float *B, float *C, int N)
{
    // Allocate shared memory for tiles of A and B
    __shared__ float tileA[32][32];
    __shared__ float tileB[32][32];

    // Calculate the row and column index for C
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0;

    // Loop over the tiles of A and B
    for (int t = 0; t < N/32; ++t) {
        // Load data into shared memory
        tileA[threadIdx.y][threadIdx.x] = A[row * N + t * 32 + threadIdx.x];
        tileB[threadIdx.y][threadIdx.x] = B[(t * 32 + threadIdx.y) * N + col];

        // Synchronize to ensure all data is loaded
        __syncthreads();

        // Perform computation on the tile
        for (int n = 0; n < 32; ++n) {
            sum += tileA[threadIdx.y][n] * tileB[n][threadIdx.x];
        }

        // Synchronize to make sure that computation is done before loading new tiles
        __syncthreads();
    }

    // Write the result to C
    if (row < N && col < N)
        C[row * N + col] = sum;
}