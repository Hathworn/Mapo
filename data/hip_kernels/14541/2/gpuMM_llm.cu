#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpuMM(float* A, float* B, float* C, int N)
{
    // Thread identifiers
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Ensure row and column are within bounds
    if (row < N && col < N) 
    {
        // Compute element C[row, col]
        float sum = 0.f;
        for (int n = 0; n < N; ++n)
        {
            sum += A[row * N + n] * B[n * N + col];
        }
        C[row * N + col] = sum;
    }
}