#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulKernelTiled(float* Md, float* Nd, float* Pd, int Width)
{
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];
    int bx = blockIdx.x, by = blockIdx.y;
    int tx = threadIdx.x, ty = threadIdx.y;

    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;
    float Pvalue = 0;

    // Loop over tiles
    for (int m = 0; m < (Width - 1) / TILE_WIDTH + 1; ++m) {
        // Load tiles into shared memory, check bounds
        if (Row < Width && m * TILE_WIDTH + tx < Width)
            Mds[ty][tx] = Md[Row * Width + m * TILE_WIDTH + tx];
        else
            Mds[ty][tx] = 0.0f;
        
        if (Col < Width && m * TILE_WIDTH + ty < Width)
            Nds[ty][tx] = Nd[(m * TILE_WIDTH + ty) * Width + Col];
        else
            Nds[ty][tx] = 0.0f;

        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k)
            Pvalue += Mds[ty][k] * Nds[k][tx];

        __syncthreads();
    }

    // Write back result only if within bounds
    if (Row < Width && Col < Width)
        Pd[Row * Width + Col] = Pvalue;
}