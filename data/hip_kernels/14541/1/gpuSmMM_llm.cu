#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void gpuSmMM(float *Ad, float *Bd, float *Cd, int dimension) {
    // Define shared memory for tiles
    __shared__ float Ads[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bds[TILE_WIDTH][TILE_WIDTH];
    
    // Calculate thread id
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;
    float Cvalue = 0.0f;  // Initialize accumulation result to 0
    
    // Iterate over tiles
    for (int m = 0; m < dimension / TILE_WIDTH; ++m) {
        // Load tiles into shared memory
        Ads[threadIdx.y][threadIdx.x] = Ad[row * dimension + (m * TILE_WIDTH + threadIdx.x)];
        Bds[threadIdx.y][threadIdx.x] = Bd[(m * TILE_WIDTH + threadIdx.y) * dimension + col];
        
        __syncthreads();  // Synchronize threads after loading tiles

        // Perform the partial multiplication for the current tile
        for (int k1 = 0; k1 < TILE_WIDTH; ++k1) {
            Cvalue += Ads[threadIdx.y][k1] * Bds[k1][threadIdx.x];
        }

        __syncthreads();  // Synchronize before loading the next tile
    }

    // Write result to C matrix
    Cd[row * dimension + col] = Cvalue;
}