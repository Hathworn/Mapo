#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int Width)
{
    // Calculate the row and column indices for Pd
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if (Row < Width && Col < Width) {  // Ensure within bounds
        float Pvalue = 0.0f;
        
        // Use shared memory for better coalesced access
        __shared__ float Mds[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Nds[BLOCK_SIZE][BLOCK_SIZE];
        
        for (int m = 0; m < Width / BLOCK_SIZE; ++m) {
            // Collaborative loading into shared memory
            Mds[threadIdx.y][threadIdx.x] = Md[Row * Width + m * BLOCK_SIZE + threadIdx.x];
            Nds[threadIdx.y][threadIdx.x] = Nd[(m * BLOCK_SIZE + threadIdx.y) * Width + Col];
            __syncthreads();

            // Compute Pvalue for this tile
            for (int k = 0; k < BLOCK_SIZE; ++k) {
                Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
            }
            __syncthreads();
        }
        Pd[Row * Width + Col] = Pvalue;
    }
}