#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    // Calculate the global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread does not exceed array bounds
    if (tid < N) {
        // Perform the addition
        c[tid] = a[tid] + b[tid];
    }
}