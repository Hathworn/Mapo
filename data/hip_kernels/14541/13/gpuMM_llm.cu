#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpuMM(float *A, float *B, float *C, int N)
{
    // Matrix multiplication for NxN matrices C=A*B
    // Each thread computes a block-tile of C using shared memory
    __shared__ float Asub[32][32]; // Shared memory for sub-matrix of A
    __shared__ float Bsub[32][32]; // Shared memory for sub-matrix of B

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.f;

    for (int tile = 0; tile < (N + 31) / 32; ++tile) {
        // Load elements into shared memory if within bounds
        if (tile * 32 + threadIdx.x < N && row < N)
            Asub[threadIdx.y][threadIdx.x] = A[row * N + tile * 32 + threadIdx.x];
        else
            Asub[threadIdx.y][threadIdx.x] = 0.0f;

        if (tile * 32 + threadIdx.y < N && col < N)
            Bsub[threadIdx.y][threadIdx.x] = B[(tile * 32 + threadIdx.y) * N + col];
        else
            Bsub[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        // Compute partial product for this tile
        for (int n = 0; n < 32; ++n)
            sum += Asub[threadIdx.y][n] * Bsub[n][threadIdx.x];

        __syncthreads();
    }

    // Store result if within bounds
    if (row < N && col < N)
        C[row * N + col] = sum;
}