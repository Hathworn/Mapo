#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_fmax(const int n, const REAL* x, const int offset_x, const int stride_x, const REAL* y, const int offset_y, const int stride_y, REAL* z, const int offset_z, const int stride_z) {
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a loop to ensure all threads within block process multiple elements if needed
    for (int idx = gid; idx < n; idx += blockDim.x * gridDim.x) {
        z[offset_z + idx * stride_z] = CAST(fmax)(x[offset_x + idx * stride_x], y[offset_y + idx * stride_y]);
    }
}