#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void ge_floor (const int sd, const int fd, const REAL* __restrict__ a, const int offset_a, const int ld_a, REAL* __restrict__ b, const int offset_b, const int ld_b) {
    // Calculate global index
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if the index is within bounds
    if (gid_0 < sd && gid_1 < fd) {
        // Perform floor operation and store the result
        int index_a = offset_a + gid_0 + gid_1 * ld_a;
        int index_b = offset_b + gid_0 + gid_1 * ld_b;
        b[index_b] = CAST(floor)(a[index_a]);
    }
}