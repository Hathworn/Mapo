#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

__global__ void uplo_equals_transp (const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, const REAL* b, const int offset_b, const int ld_b, int* eq_flag) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if indices are within bounds and perform necessary condition check.
    if (gid_0 < sd && gid_1 < sd && 
       ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1)) {
        const int ia = offset_a + gid_0 + gid_1 * ld_a;
        const int ib = offset_b + gid_1 + gid_0 * ld_b;

        // Atomic increment to ensure correct result with concurrent writes.
        if (a[ia] != b[ib]) {
            atomicAdd(eq_flag, 1);
        }
    }
}
}