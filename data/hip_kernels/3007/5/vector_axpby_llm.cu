#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

}

__global__ void vector_axpby (const int n, const REAL alpha, const REAL* __restrict__ x, const int offset_x, const int stride_x, const REAL beta, REAL* __restrict__ y, int offset_y, int stride_y) {

    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < n) {
        const int ix = offset_x + gid * stride_x;
        const int iy = offset_y + gid * stride_y;
        // Use FMA operations to improve performance by reducing rounding errors
        y[iy] = fma(alpha, x[ix], beta * y[iy]);
    }
}