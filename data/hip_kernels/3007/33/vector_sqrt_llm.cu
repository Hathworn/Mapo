#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_sqrt(const int n, const REAL* __restrict__ x, const int offset_x, const int stride_x, REAL* __restrict__ y, const int offset_y, const int stride_y) {
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < n) {
        // Reduce pointer arithmetic by computing once outside the kernel.
        const int x_index = offset_x + gid * stride_x;
        const int y_index = offset_y + gid * stride_y;
        y[y_index] = CAST(sqrt)(x[x_index]);
    }
}