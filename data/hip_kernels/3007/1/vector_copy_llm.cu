#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef NUMBER
#define NUMBER float
#endif

}

__global__ void vector_copy (const int n, const NUMBER* x, const int offset_x, const int stride_x, NUMBER* y, const int offset_y, const int stride_y) {
    // Calculate global index
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if gid is within bounds
    if (gid < n) {
        // Calculate source and destination indices
        const int ix = offset_x + gid * stride_x;
        const int iy = offset_y + gid * stride_y;
        
        // Perform copy operation
        y[iy] = x[ix];
    }
}