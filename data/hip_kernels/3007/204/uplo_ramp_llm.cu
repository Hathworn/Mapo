#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void uplo_ramp (const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Use shared memory to improve access locality
    __shared__ REAL sharedMem[256];
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    
    const bool valid = (gid_0 < sd) && (gid_1 < sd);
    const bool check = valid && ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);

    if (valid) {
        // Load data into shared memory
        sharedMem[threadIdx.x + threadIdx.y * blockDim.x] = a[offset_a + gid_0 + gid_1 * ld_a];
    }
    __syncthreads(); // Synchronize threads to ensure shared memory is fully populated

    if (check) {
        // Use shared memory for computation
        b[offset_b + gid_0 + gid_1 * ld_b] = CAST(fmax)(sharedMem[threadIdx.x + threadIdx.y * blockDim.x], (REAL)0.0);
    }
}