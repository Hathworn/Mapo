#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_div(const int n, const REAL* x, const int offset_x, const int stride_x, const REAL* y, const int offset_y, const int stride_y, REAL* z, const int offset_z, const int stride_z) {

    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Boundary check to ensure gid index is within bounds of the vector length
    if (gid < n) {
        // Prefetch indices to reduce index computation overhead
        const int x_index = offset_x + gid * stride_x;
        const int y_index = offset_y + gid * stride_y;
        const int z_index = offset_z + gid * stride_z;

        // Perform division and store result
        z[z_index] = x[x_index] / y[y_index];
    }
}