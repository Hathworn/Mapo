#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void uplo_sqr (const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Calculate global thread ID
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if thread is within matrix dimensions and if condition is satisfied
    if ((gid_0 < sd) && (gid_1 < sd) &&
        ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1)) {
        
        // Load element from matrix A, square it, and store in matrix B
        const REAL aval = a[offset_a + gid_0 + gid_1 * ld_a];
        b[offset_b + gid_0 + gid_1 * ld_b] = aval * aval;
    }
}