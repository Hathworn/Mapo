#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_exp10 (const int n, const REAL* __restrict__ x, const int offset_x, const int stride_x, REAL* __restrict__ y, const int offset_y, const int stride_y) {
    // Use shared memory for data coalescing (optional)
    extern __shared__ REAL shared_x[];
    
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < n) {
        // Use registers for better performance and avoid offset calculation in loop
        const int x_index = offset_x + gid * stride_x;
        const int y_index = offset_y + gid * stride_y;

        // Optionally use shared memory if within block size
        shared_x[threadIdx.x] = x[x_index];

        // Synchronize to ensure all numbers are loaded before computation
        __syncthreads();

        y[y_index] = CAST(exp10)(shared_x[threadIdx.x]);
    }
}