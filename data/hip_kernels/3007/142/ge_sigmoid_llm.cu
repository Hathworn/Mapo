#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void ge_sigmoid (const int sd, const int fd, const REAL* __restrict__ a, const int offset_a, const int ld_a, REAL* __restrict__ b, const int offset_b, const int ld_b) {
    // Use built-in variables and __restrict__ qualifiers to optimize memory access
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    if (gid_0 < sd && gid_1 < fd) {
        // Optimize by minimizing calculations within the loop
        REAL a_val = a[offset_a + gid_0 + gid_1 * ld_a];
        b[offset_b + gid_0 + gid_1 * ld_b] = CAST(tanh)((REAL)0.5 * a_val) * (REAL)0.5 + (REAL)0.5;
    }
}