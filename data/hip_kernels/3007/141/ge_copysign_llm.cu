#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

__global__ void ge_copysign (const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, const REAL* b, const int offset_b, const int ld_b, REAL* c, const int offset_c, const int ld_c) {
    // Calculate the global thread coordinates
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure valid thread handles valid data range
    if (gid_0 < sd && gid_1 < fd) {
        // Compute linear index for c, a, and b arrays
        const int index_a = offset_a + gid_0 + gid_1 * ld_a;
        const int index_b = offset_b + gid_0 + gid_1 * ld_b;
        const int index_c = offset_c + gid_0 + gid_1 * ld_c;
        
        // Perform copysign operation using computed indices
        c[index_c] = CAST(copysign)(a[index_a], b[index_b]);
    }
}

}