#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_log2 (const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {
    // Use shared memory to store x values to improve memory access patterns
    extern __shared__ REAL shared_x[];
    const int tid = threadIdx.x;
    const int gid = blockIdx.x * blockDim.x + tid;

    if (gid < n) {
        // Load x into shared memory
        shared_x[tid] = x[offset_x + gid * stride_x];
        __syncthreads(); // Ensure all x values are loaded

        // Write the result to y
        y[offset_y + gid * stride_y] = CAST(log2)(shared_x[tid]);
    }
}