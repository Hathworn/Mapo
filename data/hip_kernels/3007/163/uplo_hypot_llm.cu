#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

__global__ void uplo_hypot (const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, const REAL* b, const int offset_b, const int ld_b, REAL* c, const int offset_c, const int ld_c) {
    // Calculate flattened global thread indices
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Prefetch the multipliers
    const REAL multiplier_a = bottom * gid_0;
    const REAL multiplier_b = bottom * gid_1;

    // Check the conditions for valid execution
    if ((gid_0 < sd) && (gid_1 < sd) && 
        ((unit == 132) ? multiplier_a > multiplier_b : multiplier_a >= multiplier_b)) 
    {
        // Calculate the indices for input arrays and compute hypot
        const int idx_a = offset_a + gid_0 + gid_1 * ld_a;
        const int idx_b = offset_b + gid_0 + gid_1 * ld_b;
        const int idx_c = offset_c + gid_0 + gid_1 * ld_c;
        c[idx_c] = CAST(hypot)(a[idx_a], b[idx_b]);
    }
}

}