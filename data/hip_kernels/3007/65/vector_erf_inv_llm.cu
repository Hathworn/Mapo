#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_erf_inv (const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID
    if (gid >= n) return; // Early exit for threads outside the limit
    // Calculate indices and perform operation
    y[offset_y + gid * stride_y] = CAST(erfinv)(x[offset_x + gid * stride_x]);
}