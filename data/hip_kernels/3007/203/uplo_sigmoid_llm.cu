#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void uplo_sigmoid (const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within bounds and condition meets
    if (gid_0 < sd && gid_1 < sd &&
        ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1)) {
        
        // Optimized computation
        const int index_a = offset_a + gid_0 + gid_1 * ld_a;
        const int index_b = offset_b + gid_0 + gid_1 * ld_b;
        b[index_b] = CAST(tanh)((REAL)0.5 * a[index_a]) * (REAL)0.5 + (REAL)0.5;
    }
}