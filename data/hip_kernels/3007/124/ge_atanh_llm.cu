#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

__global__ void ge_atanh (const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Use 1D grid for all elements and compute global index
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_threads = gridDim.x * blockDim.x;

    for (int idx = tid; idx < sd * fd; idx += total_threads) {
        int gid_0 = idx % sd;  // Calculate gid_0 from 1D index
        int gid_1 = idx / sd;  // Calculate gid_1 from 1D index
        b[offset_b + gid_0 + gid_1 * ld_b] = CAST(atanh)(a[offset_a + gid_0 + gid_1 * ld_a]);
    }
}

}