#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void uplo_log (const int sd, const int unit, const int bottom, const REAL* __restrict__ a, const int offset_a, const int ld_a, REAL* __restrict__ b, const int offset_b, const int ld_b) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Combine conditions to determine validity only once per thread
    if ((gid_0 < sd) && (gid_1 < sd) && ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1)) {
        // Use memory coalescing to improve global memory access pattern
        int index_a = offset_a + gid_0 + gid_1 * ld_a;
        int index_b = offset_b + gid_0 + gid_1 * ld_b;
        
        b[index_b] = CAST(log)(a[index_a]);  // Use cached variables
    }
}