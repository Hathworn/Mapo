#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

__global__ void vector_elu (const int n, const REAL alpha, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {
    // Improve data locality by minimizing complex calculations in each iteration
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < n) {
        const int x_index = offset_x + gid * stride_x;
        const int y_index = offset_y + gid * stride_y;
        const REAL val = x[x_index];
        y[y_index] = CAST(fmax)(val, alpha * expm1(val)); // Calculate y in one step
    }
}

}