#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void ge_powx(const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, const REAL b, REAL* c, const int offset_c, const int ld_c) {
    // Compute global thread indices
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if ((gid_0 < sd) && (gid_1 < fd)) {
        // Compute linear indices
        int idx_a = offset_a + gid_0 + gid_1 * ld_a;
        int idx_c = offset_c + gid_0 + gid_1 * ld_c;

        // Perform power calculation and assign result
        c[idx_c] = CAST(pow)(a[idx_a], b);
    }
}