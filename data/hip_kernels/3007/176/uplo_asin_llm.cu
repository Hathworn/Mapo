#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void uplo_asin(const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    // Use shared memory to reduce global memory accesses
    __shared__ REAL shared_a[1024];

    const int tid = threadIdx.x + threadIdx.y * blockDim.x;
    if (tid < sd * sd) {
        shared_a[tid] = a[offset_a + tid];
    }
    __syncthreads();

    const bool valid = (gid_0 < sd) && (gid_1 < sd);
    const bool check = valid &&
        ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);

    if (check) {
        // Optimize using pre-loaded shared memory
        b[offset_b + gid_0 + gid_1 * ld_b] = CAST(asin)(shared_a[gid_0 + gid_1 * ld_a]);
    }
}