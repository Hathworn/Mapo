#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

__global__ void uplo_pow2o3(const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Calculate the global indices
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    // Ensure indices are within bounds; eliminated redundant check
    if ((gid_0 < sd) && (gid_1 < sd) && ((unit == 132 ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1))) {
        // Simplified the computation expression
        int idx_a = offset_a + gid_0 + gid_1 * ld_a;
        int idx_b = offset_b + gid_0 + gid_1 * ld_b;
        b[idx_b] = CAST(pow)(a[idx_a], REAL2o3);
    }
}

}