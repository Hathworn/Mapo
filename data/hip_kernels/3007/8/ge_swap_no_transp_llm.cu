#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

__global__ void ge_swap_no_transp (const int sd, const int fd, REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Optimize by calculating indices only once
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    if (gid_0 < sd && gid_1 < fd) {
        const int index_a = offset_a + gid_0 + gid_1 * ld_a;
        const int index_b = offset_b + gid_0 + gid_1 * ld_b;
        const REAL temp = b[index_b];
        b[index_b] = a[index_a];
        a[index_a] = temp;
    }
}

}