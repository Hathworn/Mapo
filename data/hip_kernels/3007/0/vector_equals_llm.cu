#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef NUMBER
#define NUMBER float
#endif

}

__global__ void vector_equals (const int n, const NUMBER* x, const int offset_x, const int stride_x, const NUMBER* y, const int offset_y, const int stride_y, int* eq_flag) {

    // Utilize shared memory to reduce atomic updates to global memory
    __shared__ int local_eq_flag;

    if(threadIdx.x == 0) {
        local_eq_flag = 0; // Initialize shared counter
    }
    __syncthreads();

    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < n) {
        const int ix = offset_x + gid * stride_x;
        const int iy = offset_y + gid * stride_y;
        if (x[ix] != y[iy]) {
            atomicAdd(&local_eq_flag, 1); // Accumulate in shared memory
        }
    }
    __syncthreads();

    // Update global counter with block's result
    if(threadIdx.x == 0) {
        atomicAdd(eq_flag, local_eq_flag);
    }
}