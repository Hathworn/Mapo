#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_log(const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {
    // Get global thread ID
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimization: Using shared memory for more efficient data access
    extern __shared__ REAL shared_x[];

    // Load data into shared memory
    if (gid < n) {
        shared_x[threadIdx.x] = x[offset_x + gid * stride_x];
        __syncthreads();

        // Apply log function and store result
        y[offset_y + gid * stride_y] = CAST(log)(shared_x[threadIdx.x]);
    }
}