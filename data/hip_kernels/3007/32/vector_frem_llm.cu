#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_frem(const int n, const REAL* x, const int offset_x, const int stride_x, const REAL* y, const int offset_y, const int stride_y, REAL* z, const int offset_z, const int stride_z) {
    // Calculate global thread ID
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within bounds
    if (gid < n) {
        // Compute remainder and store result
        z[offset_z + gid * stride_z] = CAST(remainder)(x[offset_x + gid * stride_x], y[offset_y + gid * stride_y]);
    }
}