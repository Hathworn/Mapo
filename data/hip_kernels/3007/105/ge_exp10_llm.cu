#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

__global__ void ge_exp10 (const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Use shared memory for faster data access if applicable
    __shared__ REAL shared_a_tile[32][32];
    const int lane_0 = threadIdx.x;
    const int lane_1 = threadIdx.y;
    if (gid_0 < sd && gid_1 < fd) {
        shared_a_tile[lane_0][lane_1] = a[offset_a + gid_0 + gid_1 * ld_a];
        __syncthreads(); // Ensure all threads have loaded their tiles

        // Compute exp10 using shared memory
        b[offset_b + gid_0 + gid_1 * ld_b] = CAST(exp10)(shared_a_tile[lane_0][lane_1]);
    }
}
```
