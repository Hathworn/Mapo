#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif


__global__ void ge_linear_frac (const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, const REAL* b, const int offset_b, const int ld_b, const REAL scalea, const REAL shifta, const REAL scaleb, const REAL shiftb, REAL* c, const int offset_c, const int ld_c) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Check for valid global index, assuming dimensions are multiples of block size
    if ((gid_0 < sd) && (gid_1 < fd)) {
        const int idx_a = offset_a + gid_0 + gid_1 * ld_a; // Pre-compute index for a
        const int idx_b = offset_b + gid_0 + gid_1 * ld_b; // Pre-compute index for b
        const int idx_c = offset_c + gid_0 + gid_1 * ld_c; // Pre-compute index for c

        // Simplify computation by using pre-computed indices
        c[idx_c] = (scalea * a[idx_a] + shifta) / (scaleb * b[idx_b] + shiftb);
    }
}

}