#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_cos(const int n, const REAL* __restrict__ x, const int offset_x, const int stride_x, REAL* __restrict__ y, const int offset_y, const int stride_y) {
    // Obtain the global thread ID
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the global thread ID is within the array bounds
    if (gid < n) {
        // Compute the offset for input and output vectors and apply cosine function
        const int x_index = offset_x + gid * stride_x;
        const int y_index = offset_y + gid * stride_y;
        y[y_index] = CAST(cos)(x[x_index]);
    }
}