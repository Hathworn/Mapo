#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

__global__ void ge_log10 (const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Calculate global index for each dimension
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Perform bounds check
    if (gid_0 < sd && gid_1 < fd) {
        // Perform memory access and compute in a single step
        b[offset_b + gid_0 + gid_1 * ld_b] = CAST(log10)(a[offset_a + gid_0 + gid_1 * ld_a]);
    }
}

}