#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void uplo_cdf_norm_inv (const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Compute the global thread indices
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x; 
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the indices are within bounds
    if (gid_0 < sd && gid_1 < sd) {
        // Precompute condition result to avoid repeated computation
        const bool condition_result = ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);
        
        // Apply the transformation if condition passes
        if (condition_result) {
            b[offset_b + gid_0 + gid_1 * ld_b] = CAST(normcdfinv)(a[offset_a + gid_0 + gid_1 * ld_a]);
        }
    }
}