#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

__global__ void uplo_scal (const int sd, const int unit, const int bottom, const REAL alpha, REAL* a, const int offset_a, const int ld_a) {
    // Calculate global thread indices
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Eliminate unnecessary bool checks and directly use conditions
    if (gid_0 < sd && gid_1 < sd && ((unit == 132) ? (bottom * gid_0 > bottom * gid_1) : (bottom * gid_0 >= bottom * gid_1))) {
        // Directly access and modify matrix elements
        a[offset_a + gid_0 + gid_1 * ld_a] *= alpha;
    }
}

}