#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_ramp (const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds and process data
    if (gid < n) {
        int x_index = offset_x + gid * stride_x;
        int y_index = offset_y + gid * stride_y;
        y[y_index] = CAST(fmax)(x[x_index], (REAL)0.0);
    }
}