#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_powx (const int n, const REAL* x, const int offset_x, const int stride_x, const REAL b, REAL* y, const int offset_y, const int stride_y) {

    // Calculate global index
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure gid is within bounds
    if (gid < n) {
        // Direct access and computation
        const int x_index = offset_x + gid * stride_x;
        const int y_index = offset_y + gid * stride_y;

        // Perform power calculation
        y[y_index] = CAST(pow)(x[x_index], b);
    }
}