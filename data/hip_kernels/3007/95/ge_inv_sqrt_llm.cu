#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void ge_inv_sqrt (const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the global index is within bounds
    if (gid_0 < sd && gid_1 < fd) {
        const int a_idx = offset_a + gid_0 + gid_1 * ld_a;
        const int b_idx = offset_b + gid_0 + gid_1 * ld_b;
        
        // Apply inverse square root to the element
        b[b_idx] = CAST(rsqrt)(a[a_idx]);
    }
}