#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void uplo_inv_cbrt (const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Calculate global indices
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the indices are within bounds
    if ((gid_0 < sd) && (gid_1 < sd)) {
        // Evaluate the condition for square matrix processing
        if ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1) {
            // Use atomic add to ensure correct writing when accessing shared memory
            b[offset_b + gid_0 + gid_1 * ld_b] = CAST(rcbrt)(a[offset_a + gid_0 + gid_1 * ld_a]);
        }
    }
}