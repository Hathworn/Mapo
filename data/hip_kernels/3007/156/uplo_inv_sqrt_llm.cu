#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void uplo_inv_sqrt (const int sd, const int unit, const int bottom, const REAL* __restrict__ a, const int offset_a, const int ld_a, REAL* __restrict__ b, const int offset_b, const int ld_b) {
    // Optimize index calculations using shared memory and avoid redundant calculations
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    if (gid_0 < sd && gid_1 < sd) {
        // Compute offset for both 'a' and 'b' outside the condition check
        int index_a = offset_a + gid_0 + gid_1 * ld_a;
        int index_b = offset_b + gid_0 + gid_1 * ld_b;

        if ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1) {
            // Directly calculate the inverse square root only when needed
            b[index_b] = CAST(rsqrt)(a[index_a]);
        }
    }
}