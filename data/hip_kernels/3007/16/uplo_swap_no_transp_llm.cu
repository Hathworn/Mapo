#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

}

__global__ void uplo_swap_no_transp (const int sd, const int unit, const int bottom, REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if the thread is within bounds and meets the condition
    if ((gid_0 < sd) && (gid_1 < sd) &&
        ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1)) {
        
        const int ia = offset_a + gid_0 + gid_1 * ld_a;
        const int ib = offset_b + gid_0 + gid_1 * ld_b;
        
        // Swap values of a and b at the computed indices
        const REAL c = b[ib];
        b[ib] = a[ia];
        a[ia] = c;
    }
}