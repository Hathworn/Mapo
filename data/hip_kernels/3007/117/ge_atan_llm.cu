#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

__global__ void ge_atan (const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Calculate global thread ID
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if within range
    if (gid_0 < sd && gid_1 < fd) {
        // Compute and store result
        b[offset_b + gid_0 + gid_1 * ld_b] = CAST(atan)(a[offset_a + gid_0 + gid_1 * ld_a]);
    }
}
}