#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void uplo_round (const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Calculate global indices
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Validate indices
    if (gid_0 < sd && gid_1 < sd) {
        // Check condition for processing
        if ((unit == 132 ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1)) {
            // Perform rounding and assignment
            b[offset_b + gid_0 + gid_1 * ld_b] = CAST(round)(a[offset_a + gid_0 + gid_1 * ld_a]);
        }
    }
}