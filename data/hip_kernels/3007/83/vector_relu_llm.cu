#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_relu (const int n, const REAL alpha, const REAL* __restrict__ x, const int offset_x, const int stride_x, REAL* __restrict__ y, const int offset_y, const int stride_y) {

    // Calculate global index
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use loop unrolling for better performance
    for (int idx = gid; idx < n; idx += gridDim.x * blockDim.x) {
        const REAL val = x[offset_x + idx * stride_x];
        y[offset_y + idx * stride_y] = CAST(fmax)(val, alpha * val);
    }
}