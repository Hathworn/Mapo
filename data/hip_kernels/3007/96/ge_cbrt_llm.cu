#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void ge_cbrt(const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Use shared memory for coalesced access (Assume blockDim values)
    __shared__ REAL shared_a[1024];
    if (gid_0 < sd && gid_1 < fd) {
        // Read a into shared memory
        int index_a = offset_a + gid_0 + gid_1 * ld_a;
        shared_a[threadIdx.y * blockDim.x + threadIdx.x] = a[index_a];
        __syncthreads();
        
        // Compute and store result
        int index_b = offset_b + gid_0 + gid_1 * ld_b;
        b[index_b] = CAST(cbrt)(shared_a[threadIdx.y * blockDim.x + threadIdx.x]);
    }
}