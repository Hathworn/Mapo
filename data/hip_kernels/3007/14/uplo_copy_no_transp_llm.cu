#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

}

__global__ void uplo_copy_no_transp(const int sd, const int unit, const int bottom, const REAL* __restrict__ a, const int offset_a, const int ld_a, REAL* __restrict__ b, const int offset_b, const int ld_b) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    if (gid_0 < sd && gid_1 < sd) { // Check within boundaries
        if ((unit == 132 ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1)) { // Check condition
            const int ia = offset_a + gid_0 + gid_1 * ld_a; // Calculate input index
            const int ib = offset_b + gid_0 + gid_1 * ld_b; // Calculate output index
            b[ib] = a[ia]; // Perform the copy
        }
    }
}