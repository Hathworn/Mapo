#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

__global__ void uplo_exp10 (const int sd, const int unit, const int bottom, const REAL* __restrict__ a, const int offset_a, const int ld_a, REAL* __restrict__ b, const int offset_b, const int ld_b) {
    // Utilize shared memory to reduce global memory accesses
    __shared__ REAL a_shared[blockDim.x][blockDim.y];
    
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    if (gid_0 < sd && gid_1 < sd) {
        a_shared[threadIdx.x][threadIdx.y] = a[offset_a + gid_0 + gid_1 * ld_a];
        __syncthreads();  // Ensure all threads have loaded data into shared memory
        
        if ((unit == 132 ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1)) {
            b[offset_b + gid_0 + gid_1 * ld_b] = CAST(exp10)(a_shared[threadIdx.x][threadIdx.y]);
        }
    }
}
```
