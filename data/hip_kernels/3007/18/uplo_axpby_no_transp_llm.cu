#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

}

__global__ void uplo_axpby_no_transp (const int sd, const int unit, const int bottom, const REAL alpha, const REAL* __restrict__ a, const int offset_a, const int ld_a, const REAL beta, REAL* __restrict__ b, const int offset_b, const int ld_b) {
    // Calculate global indices for the current thread
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Validate indices and condition for processing
    if ((gid_0 < sd) && (gid_1 < sd) && 
        ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1)) {

        // Compute indices for accessing arrays
        const int ia = offset_a + gid_0 + gid_1 * ld_a;
        const int ib = offset_b + gid_0 + gid_1 * ld_b;

        // Perform the axpby operation
        b[ib] = alpha * a[ia] + beta * b[ib];
    }
}