#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void ge_log2(const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Calculate global 1D index for threads
    int tid = blockIdx.x * blockDim.x + threadIdx.x + 
              (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x;
    // Check if the thread is within bounds
    if (tid < sd * fd) {
        int gid_0 = tid % sd;
        int gid_1 = tid / sd;
        b[offset_b + gid_0 + gid_1 * ld_b] = CAST(log2)(a[offset_a + gid_0 + gid_1 * ld_a]);
    }
}