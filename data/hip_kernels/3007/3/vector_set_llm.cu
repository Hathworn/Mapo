#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef NUMBER
#define NUMBER float
#endif

}

// Optimized kernel function using shared memory
__global__ void vector_set(const int n, const NUMBER val, NUMBER* x, const int offset_x, const int stride_x) {
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure gid is not computed multiple times if n is small
    if (gid < n) {
        x[offset_x + gid * stride_x] = val;
    }
}