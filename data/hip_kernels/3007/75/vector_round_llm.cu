#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_round (const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {

    // Calculate global index
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure calculation only if gid is within bounds
    if (gid < n) {
        // Fetch input and write rounded output to global memory
        REAL val = x[offset_x + gid * stride_x];
        y[offset_y + gid * stride_y] = (REAL)CAST(lrint)(val);
    }
}