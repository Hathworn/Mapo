#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_pow3o2(const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {
    // Calculate global index
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    // Check boundary condition
    if (gid >= n) return;
    // Compute power 3/2 and store the result
    y[offset_y + gid * stride_y] = CAST(pow)(x[offset_x + gid * stride_x], REAL3o2);
}