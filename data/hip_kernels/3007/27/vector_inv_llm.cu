#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_inv (const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {

    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for potential reuse and lower global memory traffic
    __shared__ REAL shared_x[256];  // Assuming blockDim.x <= 256

    // Load data into shared memory
    if (gid < n) {
        shared_x[threadIdx.x] = x[offset_x + gid * stride_x];
        __syncthreads();
        
        // Use shared memory for further computation
        y[offset_y + gid * stride_y] = (REAL)1.0 / shared_x[threadIdx.x];
    }
}