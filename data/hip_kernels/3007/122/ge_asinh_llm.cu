#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}
__global__ void ge_asinh (const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Calculate global index once and store in register for reuse
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    // Check if the thread index is within bounds
    if (gid_0 < sd && gid_1 < fd) {
        // Calculate flattened index to access global memory efficiently
        const int idx_a = offset_a + gid_0 + gid_1 * ld_a;
        const int idx_b = offset_b + gid_0 + gid_1 * ld_b;
        // Directly save result, minimizing memory accesses
        b[idx_b] = CAST(asinh)(a[idx_a]);
    }
}