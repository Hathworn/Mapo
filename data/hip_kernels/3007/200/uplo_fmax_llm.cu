#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

__global__ void uplo_fmax (const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, const REAL* b, const int offset_b, const int ld_b, REAL* c, const int offset_c, const int ld_c) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    if (gid_0 >= sd || gid_1 >= sd) return; // Early exit for invalid threads

    bool is_valid = (unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1;
    
    if (is_valid) {
        int a_index = offset_a + gid_0 + gid_1 * ld_a;
        int b_index = offset_b + gid_0 + gid_1 * ld_b;
        int c_index = offset_c + gid_0 + gid_1 * ld_c;

        // Perform fmax and store the result in c
        c[c_index] = CAST(fmax)(a[a_index], b[b_index]);
    }
}
}