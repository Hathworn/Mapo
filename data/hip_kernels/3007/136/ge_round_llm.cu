#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

__global__ void ge_round (const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    if (gid_0 < sd && gid_1 < fd) {  // Combined validity check into the conditional statement
        // Calculate input and output indices outside of the loop
        int input_index = offset_a + gid_0 + gid_1 * ld_a;
        int output_index = offset_b + gid_0 + gid_1 * ld_b;
        b[output_index] = CAST(round)(a[input_index]);
    }
}
}