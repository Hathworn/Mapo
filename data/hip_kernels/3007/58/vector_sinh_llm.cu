#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_sinh (const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {

    // Calculate global thread ID
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to optimize global memory access
    extern __shared__ REAL shared_x[];
    
    if (gid < n) {
        // Load data into shared memory
        shared_x[threadIdx.x] = x[offset_x + gid * stride_x];
        __syncthreads();

        // Perform computation using shared memory
        y[offset_y + gid * stride_y] = CAST(sinh)(shared_x[threadIdx.x]);
    }
}