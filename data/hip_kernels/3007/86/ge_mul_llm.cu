#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void ge_mul (const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, const REAL* b, const int offset_b, const int ld_b, REAL* c, const int offset_c, const int ld_c) {
    // Calculate global thread indices
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Combine index calculations to reduce operations
    const int index_a = offset_a + gid_0 + gid_1 * ld_a;
    const int index_b = offset_b + gid_0 + gid_1 * ld_b;
    const int index_c = offset_c + gid_0 + gid_1 * ld_c;

    // Execute multiplication only for valid indices
    if (gid_0 < sd && gid_1 < fd) {
        c[index_c] = a[index_a] * b[index_b];
    }
}