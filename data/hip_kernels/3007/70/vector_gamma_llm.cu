#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_gamma (const int n, const REAL* __restrict__ x, const int offset_x, const int stride_x, REAL* __restrict__ y, const int offset_y, const int stride_y) {
    // Calculate global thread ID
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // In-bounds check to avoid out-of-bounds memory access
    if (gid < n) {
        // Perform computation with optimized memory access
        y[offset_y + gid * stride_y] = CAST(tgamma)(x[offset_x + gid * stride_x]);
    }
}