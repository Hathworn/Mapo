#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

}

__global__ void ge_swap_transp(const int sd, const int fd, REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Calculate global indices
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if indices are within bounds
    if (gid_0 < sd && gid_1 < fd) {
        // Calculate linear indices for matrices `a` and `b`
        const int ia = offset_a + gid_0 + gid_1 * ld_a;
        const int ib = offset_b + gid_1 + gid_0 * ld_b;
        
        // Perform the transposition and swap
        REAL c = b[ib]; // Load from b
        b[ib] = a[ia];  // Store to b
        a[ia] = c;      // Store to a
    }
}