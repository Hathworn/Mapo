#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void uplo_copysign (const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, const REAL* b, const int offset_b, const int ld_b, REAL* c, const int offset_c, const int ld_c) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure threads only proceed if they are within bounds
    if (gid_0 < sd && gid_1 < sd) {
        // Pre-compute indices for memory access
        int idx_a = offset_a + gid_0 + gid_1 * ld_a;
        int idx_b = offset_b + gid_0 + gid_1 * ld_b;
        int idx_c = offset_c + gid_0 + gid_1 * ld_c;

        // Calculate condition for unit type
        bool condition = (unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1;

        // Perform copysign operation if condition is met
        if (condition) {
            c[idx_c] = CAST(copysign)(a[idx_a], b[idx_b]);
        }
    }
}