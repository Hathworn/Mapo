#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_atanh (const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {

    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a loop to handle potential out-of-bounds accesses in case of non-multiple block/thread sizes
    for(int i = gid; i < n; i += gridDim.x * blockDim.x) {
        y[offset_y + i * stride_y] = CAST(atanh)(x[offset_x + i * stride_x]);
    }
}