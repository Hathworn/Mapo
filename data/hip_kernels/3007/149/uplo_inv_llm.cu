#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void uplo_inv (const int sd, const int unit, const int bottom, const REAL* __restrict__ a, const int offset_a, const int ld_a, REAL* __restrict__ b, const int offset_b, const int ld_b) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    if (gid_0 < sd && gid_1 < sd) {  // Inline validity check for efficiency
        const int index_a = offset_a + gid_0 + gid_1 * ld_a;  // Compute a index once
        const int index_b = offset_b + gid_0 + gid_1 * ld_b;  // Compute b index once
        // Pre-calculate multiplication to avoid repetitive calculations
        const int comp_g0_g1 = bottom * gid_0;
        const int comp_g1_g0 = bottom * gid_1;
        if ((unit == 132 ? comp_g0_g1 > comp_g1_g0 : comp_g0_g1 >= comp_g1_g0)) {
            b[index_b] = (REAL)1.0 / a[index_a];  // Perform division
        }
    }
}