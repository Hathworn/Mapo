#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void uplo_linear_frac (const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, const REAL* b, const int offset_b, const int ld_b, const REAL scalea, const REAL shifta, const REAL scaleb, const REAL shiftb, REAL* c, const int offset_c, const int ld_c) {
    // Compute global indices
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if threads are within bounds
    if ((gid_0 < sd) && (gid_1 < sd)) {
        // Determine if computation is necessary based on 'unit' and 'bottom' values
        bool condition = (unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1;
        
        // Perform computation if condition is met
        if (condition) {
            int idx = gid_0 + gid_1 * ld_a; // Compute index once for reuse
            c[offset_c + gid_0 + gid_1 * ld_c] =
                (scalea * a[offset_a + idx] + shifta) /
                (scaleb * b[offset_b + idx] + shiftb);
        }
    }
}