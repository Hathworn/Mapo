#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

}

__global__ void uplo_copy_transp(const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Using shared memory to minimize global memory accesses
    __shared__ REAL tile[32][32+1]; // +1 to avoid bank conflicts

    int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    if (gid_0 < sd && gid_1 < sd) {
        tile[threadIdx.y][threadIdx.x] = a[offset_a + gid_0 + gid_1 * ld_a];
    }
    __syncthreads();

    gid_0 = blockIdx.y * blockDim.y + threadIdx.x;
    gid_1 = blockIdx.x * blockDim.x + threadIdx.y;

    if (gid_0 < sd && gid_1 < sd) {
        bool check = (unit == 132) ? bottom * gid_1 > bottom * gid_0 : bottom * gid_1 >= bottom * gid_0;
        if (check) {
            b[offset_b + gid_1 + gid_0 * ld_b] = tile[threadIdx.x][threadIdx.y];
        }
    }
}