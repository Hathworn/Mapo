#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void ge_tan(const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Calculate global index for current thread
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if thread is within bounds
    if (gid_0 < sd && gid_1 < fd) {
        // Use shared memory to store intermediate values
        extern __shared__ REAL shared_a[];
        const int local_idx = threadIdx.y * blockDim.x + threadIdx.x;
        shared_a[local_idx] = a[offset_a + gid_0 + gid_1 * ld_a];

        // Synchronize threads in the block
        __syncthreads();

        // Write result
        b[offset_b + gid_0 + gid_1 * ld_b] = CAST(tan)(shared_a[local_idx]);
    }
}