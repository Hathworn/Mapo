#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

__global__ void vector_atan2(const int n, const REAL* x, const int offset_x, const int stride_x, 
                             const REAL* y, const int offset_y, const int stride_y, 
                             REAL* z, const int offset_z, const int stride_z) {
    // Calculate global thread index
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread is within bounds
    if (gid < n) {
        // Compute and store result
        int idx_x = offset_x + gid * stride_x;
        int idx_y = offset_y + gid * stride_y;
        int idx_z = offset_z + gid * stride_z;
        z[idx_z] = CAST(atan2)(x[idx_x], y[idx_y]);
    }
}

}