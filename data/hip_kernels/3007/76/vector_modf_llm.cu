#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_modf (const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y, REAL* z, const int offset_z, const int stride_z) {

    // Calculate global thread ID
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread ID is within bounds
    if (gid < n) {
        const int x_idx = offset_x + gid * stride_x;
        const int y_idx = offset_y + gid * stride_y;
        const int z_idx = offset_z + gid * stride_z;
        
        // Optimized memory access
        REAL x_value = x[x_idx];
        z[z_idx] = CAST(modf)(x_value, &y[y_idx]);
    }
}