#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

}

__global__ void ge_set(const int sd, const int fd, const REAL val, REAL* a, const int offset_a, const int ld_a) {
    // Calculate global thread ID for 1D flattened grid
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate offsets in 2D grid
    int gid_x = gid % sd;
    int gid_y = gid / sd;
    
    // Check if global thread coordinates are within matrix bounds
    if (gid_x < sd && gid_y < fd) {
        // Set the value in the matrix at calculated position
        a[offset_a + gid_x + gid_y * ld_a] = val;
    }
}