#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_hypot (const int n, const REAL* x, const int offset_x, const int stride_x, const REAL* y, const int offset_y, const int stride_y, REAL* z, const int offset_z, const int stride_z) {

    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure gid is within the bounds before computation
    if (gid >= n) return;

    // Compute the index once and reuse it
    const int index = gid * stride_z + offset_z;
    
    // Optimize memory access by using the computed index
    z[index] = CAST(hypot)(
                   x[offset_x + gid * stride_x], 
                   y[offset_y + gid * stride_y]);
}