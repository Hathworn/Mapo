#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_cdf_norm_inv (const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {
    // Calculate global ID once and store in a register
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use conditional gating to prevent out-of-bounds access
    if (gid < n) {
        const int x_index = offset_x + gid * stride_x; // Pre-calculate memory index
        const int y_index = offset_y + gid * stride_y; // Pre-calculate memory index
        y[y_index] = CAST(normcdfinv)(x[x_index]);
    }
}