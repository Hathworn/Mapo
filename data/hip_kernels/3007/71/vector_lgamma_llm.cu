#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_lgamma (const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a loop to process multiple elements per thread
    for (int i = gid; i < n; i += blockDim.x * gridDim.x) {
        y[offset_y + i * stride_y] = CAST(lgamma)(x[offset_x + i * stride_x]);
    }
}