#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifndef REAL
#define REAL float
#endif

__global__ void uplo_equals_no_transp(const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, const REAL* b, const int offset_b, const int ld_b, int* eq_flag) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread indices are within matrix dimensions
    if (gid_0 < sd && gid_1 < sd) {
        const int comparison = (unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1;
        
        // Comparison to determine whether to proceed with element comparison
        if (comparison) {
            const int ia = offset_a + gid_0 + gid_1 * ld_a;
            const int ib = offset_b + gid_0 + gid_1 * ld_b;
            
            // Atomic operation to safely update eq_flag in parallel
            if (a[ia] != b[ib]) {
                atomicAdd(eq_flag, 1);
            }
        }
    }
}