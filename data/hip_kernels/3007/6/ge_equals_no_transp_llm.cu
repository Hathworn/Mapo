#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

__global__ void ge_equals_no_transp (const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, const REAL* b, const int offset_b, const int ld_b, int* eq_flag) {
    // Compute global thread indices
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if indices are within valid range
    if ((gid_0 < sd) && (gid_1 < fd)) {
        // Compute flattened array indices
        const int ia = offset_a + gid_0 + gid_1 * ld_a;
        const int ib = offset_b + gid_0 + gid_1 * ld_b;

        // Check for inequality and accumulate flag atomically
        if (a[ia] != b[ib]) {
            atomicAdd(eq_flag, 1);  // Use atomic add for concurrent access
        }
    }
}
}