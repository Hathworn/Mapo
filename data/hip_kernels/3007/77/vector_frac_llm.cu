#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_frac (const int n, const REAL* __restrict__ x, const int offset_x, const int stride_x, REAL* __restrict__ y, const int offset_y, const int stride_y) {

    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    if (gid < n) {
        REAL dummy;
        // Optimized: Use local computation to reduce global memory access
        const REAL val = x[offset_x + gid * stride_x];
        y[offset_y + gid * stride_y] = CAST(modf)(val, &dummy);
    }
}