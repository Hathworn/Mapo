#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void ge_sqrt (const int sd, const int fd, const REAL* __restrict__ a, const int offset_a, const int ld_a, REAL* __restrict__ b, const int offset_b, const int ld_b) {
    // Calculate unique global index for each thread
    int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure within bounds before processing
    if (gid_0 < sd && gid_1 < fd) {
        // Perform square root using the casted function
        b[offset_b + gid_0 + gid_1 * ld_b] = CAST(sqrt)(a[offset_a + gid_0 + gid_1 * ld_a]);
    }
}