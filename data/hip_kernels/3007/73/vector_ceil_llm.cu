#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_ceil(const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate grid-stride for efficient iteration

    // Use grid-stride loop to cover all elements
    for (int i = gid; i < n; i += stride) {
        y[offset_y + i * stride_y] = CAST(ceil)(x[offset_x + i * stride_x]);
    }
}