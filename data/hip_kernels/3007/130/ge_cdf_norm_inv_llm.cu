#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void ge_cdf_norm_inv(const int sd, const int fd, const REAL* __restrict__ a, const int offset_a, const int ld_a, REAL* __restrict__ b, const int offset_b, const int ld_b) {
    // Calculate global thread indices
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Optimization: use shared memory for performance improvement if needed.

    // Check bounds and perform calculation if valid
    if (gid_0 < sd && gid_1 < fd) {
        // Use efficient memory access by calculating global index once
        const int index_a = offset_a + gid_0 + gid_1 * ld_a;
        const int index_b = offset_b + gid_0 + gid_1 * ld_b;
        // Apply the normcdfinv function
        b[index_b] = CAST(normcdfinv)(a[index_a]);
    }
}