#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

__global__ void vector_mul (const int n, const REAL* x, const int offset_x, const int stride_x, const REAL* y, const int offset_y, const int stride_y, REAL* z, const int offset_z, const int stride_z) {
    // Calculate the global thread ID
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Only access valid indices within array bounds
    if (gid < n) {
        // Perform element-wise multiplication
        int x_index = offset_x + gid * stride_x;
        int y_index = offset_y + gid * stride_y;
        int z_index = offset_z + gid * stride_z;
        
        z[z_index] = x[x_index] * y[y_index];
    }
}

}