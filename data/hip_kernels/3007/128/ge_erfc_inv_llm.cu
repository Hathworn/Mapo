#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void ge_erfc_inv(const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    // Pre-calculate indices outside the conditional for better performance
    const int index_a = offset_a + gid_0 + gid_1 * ld_a;
    const int index_b = offset_b + gid_0 + gid_1 * ld_b;
    // Optimize boundary condition check
    if (gid_0 < sd && gid_1 < fd) {
        b[index_b] = CAST(erfcinv)(a[index_a]);
    }
}