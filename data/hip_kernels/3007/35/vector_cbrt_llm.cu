#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_cbrt (const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {

    // Calculate global index
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure gid is within array bounds
    if (gid < n) {
        // Efficient memory access pattern
        const int input_idx = offset_x + gid * stride_x;
        const int output_idx = offset_y + gid * stride_y;

        // Perform cubic root on input and store in output
        y[output_idx] = CAST(cbrt)(x[input_idx]);
    }
}