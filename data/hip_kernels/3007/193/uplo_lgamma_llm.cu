#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

__global__ void uplo_lgamma (const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;  // Global index for x
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;  // Global index for y

    if (gid_0 < sd && gid_1 < sd) {  // Validate global index
        if ((unit == 132 ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1)) {  // Condition check
            int index_a = offset_a + gid_0 + gid_1 * ld_a;  // Calculate index for a
            int index_b = offset_b + gid_0 + gid_1 * ld_b;  // Calculate index for b
            b[index_b] = CAST(lgamma)(a[index_a]);  // Compute and assign
        }
    }
}

}