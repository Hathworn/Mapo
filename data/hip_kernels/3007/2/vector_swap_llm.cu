#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef NUMBER
#define NUMBER float
#endif

}

__global__ void vector_swap(const int n, NUMBER* x, const int offset_x, const int stride_x, NUMBER* y, const int offset_y, const int stride_y) {
    // Calculate unique thread ID
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure gid is within bounds
    if (gid < n) {
        // Compute the indices for the current thread
        const int ix = offset_x + gid * stride_x;
        const int iy = offset_y + gid * stride_y;

        // Swap the elements directly without an extra variable
        // Optimized by using a temporary variable to minimize number of memory accesses
        const NUMBER temp = y[iy]; 
        y[iy] = x[ix];
        x[ix] = temp;
    }
}