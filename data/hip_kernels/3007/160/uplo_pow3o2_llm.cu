#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void uplo_pow3o2 (const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    if (gid_0 < sd && gid_1 < sd) { // Combine checks and reduce branching
        bool condition = (unit == 132) ? (bottom * gid_0 > bottom * gid_1) : (bottom * gid_0 >= bottom * gid_1);
        if (condition) {
            int idx_a = offset_a + gid_0 + gid_1 * ld_a; // Calculate indexes once
            int idx_b = offset_b + gid_0 + gid_1 * ld_b;
            b[idx_b] = CAST(pow)(a[idx_a], REAL3o2);
        }
    }
}