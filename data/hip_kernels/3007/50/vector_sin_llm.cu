#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_sin (const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {

    // Calculate global thread ID
    const int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Process only if within bounds
    if (gid < n) {
        // Optimize memory access patterns
        const int x_index = offset_x + gid * stride_x;
        const int y_index = offset_y + gid * stride_y;

        // Compute and store the result
        y[y_index] = CAST(sin)(x[x_index]);
    }
}