#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void ge_fmax (const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, const REAL* b, const int offset_b, const int ld_b, REAL* c, const int offset_c, const int ld_c) {
    // Calculate global indices using block and thread indices
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if the indices are valid
    if (gid_0 < sd && gid_1 < fd) {
        // Use computed indices to perform the maximum operation
        int idx_a = offset_a + gid_0 + gid_1 * ld_a;
        int idx_b = offset_b + gid_0 + gid_1 * ld_b;
        int idx_c = offset_c + gid_0 + gid_1 * ld_c;
        c[idx_c] = CAST(fmax)(a[idx_a], b[idx_b]);
    }
}