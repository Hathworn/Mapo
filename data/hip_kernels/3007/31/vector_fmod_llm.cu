#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_fmod (const int n, const REAL* __restrict__ x, const int offset_x, const int stride_x, const REAL* __restrict__ y, const int offset_y, const int stride_y, REAL* __restrict__ z, const int offset_z, const int stride_z) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop unrolling for better performance
    for (; gid < n; gid += blockDim.x * gridDim.x) {
        z[offset_z + gid * stride_z] = CAST(fmod)(x[offset_x + gid * stride_x], y[offset_y + gid * stride_y]);
    }
}