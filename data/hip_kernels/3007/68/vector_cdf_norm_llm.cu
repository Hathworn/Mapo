#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void vector_cdf_norm (const int n, const REAL* x, const int offset_x, const int stride_x, REAL* y, const int offset_y, const int stride_y) {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    /* Check if gid is within bounds and optimize memory access pattern */
    if (gid < n) {
        int x_index = offset_x + gid * stride_x;  // Precompute indices for reuse
        int y_index = offset_y + gid * stride_y;
        y[y_index] = CAST(normcdf)(x[x_index]);
    }
}