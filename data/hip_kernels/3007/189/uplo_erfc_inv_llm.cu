#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void uplo_erfc_inv(const int sd, const int unit, const int bottom, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Calculate global thread indices
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check for valid thread indices within matrix bounds
    const bool valid = (gid_0 < sd) && (gid_1 < sd);
    
    // Determine if the thread should process based on unit and bottom values
    const bool check = valid &&
    ((unit == 132) ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);
    
    // Perform the main computation if check passes
    if (check) {
        // Use optimized memory access by calculating destination directly
        const int index_a = offset_a + gid_0 + gid_1 * ld_a;
        const int index_b = offset_b + gid_0 + gid_1 * ld_b;
        b[index_b] = CAST(erfcinv)(a[index_a]);
    }
}