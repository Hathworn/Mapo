#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

#ifndef CAST
#define CAST(fun) fun ## f
#endif

#ifndef REAL2o3
#define REAL2o3 (REAL)0.6666666666666667
#endif

#ifndef REAL3o2
#define REAL3o2 (REAL)1.5
#endif

}

__global__ void ge_sin (const int sd, const int fd, const REAL* a, const int offset_a, const int ld_a, REAL* b, const int offset_b, const int ld_b) {
    // Calculate global thread indices
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within bounds
    if (gid_0 < sd && gid_1 < fd) {
        // Calculate linear index for input and output
        const int index_a = offset_a + gid_0 + gid_1 * ld_a;
        const int index_b = offset_b + gid_0 + gid_1 * ld_b;

        // Perform the operation and store result
        b[index_b] = CAST(sin)(a[index_a]);
    }
}