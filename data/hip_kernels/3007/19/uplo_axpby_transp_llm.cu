#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {

#ifndef REAL
#define REAL float
#endif

}

__global__ void uplo_axpby_transp (const int sd, const int unit, const int bottom, const REAL alpha, const REAL* a, const int offset_a, const int ld_a, const REAL beta, REAL* b, const int offset_b, const int ld_b) {
    const int gid_0 = blockIdx.x * blockDim.x + threadIdx.x;
    const int gid_1 = blockIdx.y * blockDim.y + threadIdx.y;

    // Prefetch condition results to reduce redundant calculations
    const bool valid_gid_0 = gid_0 < sd;
    const bool valid_gid_1 = gid_1 < sd;
    const bool valid = valid_gid_0 && valid_gid_1;
    const bool check = valid && (unit == 132 ? bottom * gid_0 > bottom * gid_1 : bottom * gid_0 >= bottom * gid_1);

    if (check) {
        const int ia = offset_a + gid_0 + gid_1 * ld_a;
        const int ib = offset_b + gid_1 + gid_0 * ld_b;
        // Use registers for temporary variables to optimize memory access
        const REAL a_val = a[ia];
        const REAL b_val = b[ib];
        b[ib] = alpha * a_val + beta * b_val;
    }
}