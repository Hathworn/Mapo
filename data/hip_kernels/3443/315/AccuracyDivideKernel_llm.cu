#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use block and thread index to ensure each thread processes one element if required
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        *accuracy /= N;  // Only one thread performs the division
    }
}