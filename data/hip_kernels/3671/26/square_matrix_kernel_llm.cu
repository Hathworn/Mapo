#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Use shared memory for faster data access
    __shared__ float block_feats[32][32];  // Assuming block size 32x32

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < num_rows && j < num_cols) {
        // Load data into shared memory
        block_feats[threadIdx.y][threadIdx.x] = feats[i * ldf + j];
        __syncthreads();  // Ensure all threads have loaded their data

        // Compute and store result
        float f = block_feats[threadIdx.y][threadIdx.x];
        feats_sq[i * lds + j] = f * f;
    }
}