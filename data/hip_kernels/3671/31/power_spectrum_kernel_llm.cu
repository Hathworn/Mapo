#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    // Calculate thread and block index
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Calculate pointers with block and thread index
    float *Ar = A_in + blockIdx.x * ldi;
    float *Aw = A_out + blockIdx.x * ldo;
    
    int half_length = row_length / 2;
    
    // Optimize loop by using single thread block instead of CU1DBLOCK
    for (int idx = thread_id; idx < half_length; idx += blockDim.x * gridDim.x) {
        // Ignore special case
        if (idx == 0) continue;
        
        // Transform and process computation of the complex number
        float2 val = reinterpret_cast<float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        
        // Conditional assignment for Aw array
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }
    
    // Process special case within the thread block
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];
        
        // Conditional assignment for special indices
        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}