#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Gaussian blur CUDA kernel
__global__ void GaussianBlurCuda(unsigned char *pic, unsigned char *outpic, double *mask, int *size) {
    // Calculate pixel position
    int pxPosCen = blockIdx.x * blockDim.x + threadIdx.x;
    if (pxPosCen >= size[0] * size[1]) return;

    int row = pxPosCen / size[0];
    int col = pxPosCen % size[0];
    
    // Border check
    if (row < size[2] / 2 || row >= (size[1] - size[2] / 2)) return;
    if (col < size[2] / 2 || col >= (size[0] - size[2] / 2)) return;

    double sumout[3] = {0.0, 0.0, 0.0};

    // Compute Gaussian blur
    for (int i = 0; i < size[2]; i++) {    // Loop over the mask's height
        for (int j = 0; j < size[2]; j++) { // Loop over the mask's width
            int x = j + col - size[2] / 2;
            int y = i + row - size[2] / 2;
            int pos = (y * size[0] + x) * 3;

            // Accumulate sum for RGB channels
            sumout[0] += static_cast<double>(pic[pos]) * mask[i * size[2] + j];
            sumout[1] += static_cast<double>(pic[pos + 1]) * mask[i * size[2] + j];
            sumout[2] += static_cast<double>(pic[pos + 2]) * mask[i * size[2] + j];
        }
    }

    // Write back output pixels
    int pos = pxPosCen * 3;
    outpic[pos] = static_cast<unsigned char>(sumout[0]);
    outpic[pos + 1] = static_cast<unsigned char>(sumout[1]);
    outpic[pos + 2] = static_cast<unsigned char>(sumout[2]);
}