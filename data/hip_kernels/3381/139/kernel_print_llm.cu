#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_print(int const* p, int n)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate flattened index
    if (idx < n) {
        printf("int: %d %d\n", n, p[idx]); // Use index to access array element and reduce iteration complexity
    }
}