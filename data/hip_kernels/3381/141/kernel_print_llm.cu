#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_print(size_t const* p, int n)
{
    // Cache thread index to reduce repeated calls
    int tid = threadIdx.x;
    if(tid < n) {
        // Minimize use of printf, use a single call for both values
        printf("ulong: %d %lu\n", n, p[tid]);
    }
}