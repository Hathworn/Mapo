#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void kernel_print( long const* p, int n)
{
    printf("long: %d ", n);
    
    // Loop unrolling for better performance
    int i = threadIdx.x;
    int stride = blockDim.x;
    for(; i < n; i += stride)
    {
        printf("%ld ", p[i]);
    }
}