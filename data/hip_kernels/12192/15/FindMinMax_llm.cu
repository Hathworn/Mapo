#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    const int b = blockDim.x;
    int p = y * pitch + x;
    
    // Initialize shared memory with boundary check
    float val = (x < width) ? d_Data[p] : d_Data[p - x];
    minvals[tx] = val;
    maxvals[tx] = val;

    // Loop unrolling for processing 16 consecutive rows
    for (int ty = 1; ty < 16; ty++) {
        p += pitch;
        if (tx < width && p < width * height) {
            float val = d_Data[p];
            // Min/max reduction
            if (val < minvals[tx])
                minvals[tx] = val;
            if (val > maxvals[tx])
                maxvals[tx] = val;
        }
    }
    __syncthreads();

    // Parallel reduction with loop unrolling
    for (int d = b / 2; d > 0; d >>= 1) {
        if (tx < d) {
            // Avoid unnecessary checks
            if (minvals[tx + d] < minvals[tx])
                minvals[tx] = minvals[tx + d];
            if (maxvals[tx + d] > maxvals[tx])
                maxvals[tx] = maxvals[tx + d];
        }
        __syncthreads();
    }
    
    // Store final results for each block
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
    __syncthreads();
}