#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Use built-in functions for index calculation and thread synchronization
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int p = y * width + x;  // Simplified multiplication
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
    // Remove unnecessary syncthreads
}