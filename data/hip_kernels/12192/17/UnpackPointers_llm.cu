#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;
    
    // Calculate starting position for each thread.
    int block_x = blockIdx.x * 16 + tx;
    
    // Loop through y positions in parallel with stride of the number of blocks
    for (int y = blockIdx.y; y < h / 32; y += gridDim.y) {
        // Loop through x positions in parallel, each thread handles a column
        if (block_x < w) {
            unsigned int val = minmax[y * w + block_x];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + block_x;
                        numPts++;
                    }
                    val >>= 1; // Shift right using the current thread
                }
            }
        }
    }
}