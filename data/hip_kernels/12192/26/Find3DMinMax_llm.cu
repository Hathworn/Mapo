#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int height)
{
    // Shared memory for caching data
    __shared__ float data1[3*(MINMAX_SIZE + 2)];
    __shared__ float data2[3*(MINMAX_SIZE + 2)];
    __shared__ float data3[3*(MINMAX_SIZE + 2)];
    __shared__ float ymin1[(MINMAX_SIZE + 2)];
    __shared__ float ymin2[(MINMAX_SIZE + 2)];
    __shared__ float ymin3[(MINMAX_SIZE + 2)];
    __shared__ float ymax1[(MINMAX_SIZE + 2)];
    __shared__ float ymax2[(MINMAX_SIZE + 2)];
    __shared__ float ymax3[(MINMAX_SIZE + 2)];

    // Setup thread index and memory positions
    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_SIZE;
    const int xEnd = xStart + MINMAX_SIZE - 1;
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;
    const int xEndClamped = min(xEnd, width - 1);
    int memWid = MINMAX_SIZE + 2;

    int memPos0 = (tx - WARP_SIZE + 1);
    int memPos1 = (tx - WARP_SIZE + 1);
    int yq = 0;
    unsigned int output = 0;

    for (int y = 0; y < 34; y++) {
        output >>= 1;
        int memPos = yq * memWid + (tx - WARP_SIZE + 1);
        int yp = max(min(32 * blockIdx.y + y - 1, height - 1), 0);
        int readStart = yp * width;

        // Load data into shared memory
        if (tx >= (WARP_SIZE - 1)) {
            if (xReadPos < 0 || xReadPos >= width) {
                data1[memPos] = 0;
                data2[memPos] = 0;
                data3[memPos] = 0;
            } else {
                data1[memPos] = d_Data1[readStart + xReadPos];
                data2[memPos] = d_Data2[readStart + xReadPos];
                data3[memPos] = d_Data3[readStart + xReadPos];
            }
        }
        __syncthreads();

        // Compute minima and maxima
        int memPos2 = yq * memWid + tx;
        if (y > 1 && tx < memWid) {
            float min1 = fminf(fminf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float min2 = fminf(fminf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float min3 = fminf(fminf(data3[memPos0], data3[memPos1]), data3[memPos2]);
            float max1 = fmaxf(fmaxf(data1[memPos0], data1[memPos1]), data1[memPos2]);
            float max2 = fmaxf(fmaxf(data2[memPos0], data2[memPos1]), data2[memPos2]);
            float max3 = fmaxf(fmaxf(data3[memPos0], data3[memPos1]), data3[memPos2]);

            ymin1[tx] = min1;
            ymin2[tx] = fminf(fminf(min1, min2), min3);
            ymin3[tx] = min3;
            ymax1[tx] = max1;
            ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
            ymax3[tx] = max3;
        }
        __syncthreads();

        // Determine output condition
        if (y > 1 && tx < MINMAX_SIZE && xWritePos <= xEndClamped) {
            float minv = fminf(fminf(fminf(fminf(fminf(ymin2[tx], ymin2[tx + 2]),
                ymin1[tx + 1]), ymin3[tx + 1]), data2[memPos0 + 1]), data2[memPos2 + 1]);
            minv = fminf(minv, d_ConstantA[1]);
            float maxv = fmaxf(fmaxf(fmaxf(fmaxf(fmaxf(ymax2[tx], ymax2[tx + 2]),
                ymax1[tx + 1]), ymax3[tx + 1]), data2[memPos0 + 1]), data2[memPos2 + 1]);
            maxv = fmaxf(maxv, d_ConstantA[0]);
            if (data2[memPos1 + 1] < minv || data2[memPos1 + 1] > maxv)
                output |= 0x80000000;
        }
        __syncthreads();

        // Rotate memory positions for next iteration
        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }
    if (tx < MINMAX_SIZE && xWritePos < width) {
        int writeStart = blockIdx.y * width + xWritePos;
        d_Result[writeStart] = output;
    }
}