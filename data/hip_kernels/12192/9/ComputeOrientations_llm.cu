#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOrientations(float *g_Data, float *d_Sift, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize histogram
    for (int i = 0; i < 13; i++)
        hist[i * 32 + tx] = 0.0f;
    __syncthreads();

    // Precompute Gauss values
    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15)
        gauss[tx] = exp(i2sigma2 * (tx - 7)*(tx - 7));
    
    int xp = (int)(d_Sift[bx + 0 * maxPts] - 6.5f);
    int yp = (int)(d_Sift[bx + 1 * maxPts] - 6.5f);
    int px = xp & 15;
    int x = tx - px;

    // Load data into shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = xp + x;
        int yi = yp + y;
        xi = max(0, min(w - 1, xi)); // Clamping to image boundaries
        yi = max(0, min(h - 1, yi));

        if (x >= 0 && x < 15)
            data[memPos] = g_Data[yi * w + xi];
    }
    __syncthreads();

    // Compute gradients and populate histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = 16.0f * atan2f(dy, dx) / 3.1416f + 16.5f;
            bin = (bin == 32) ? 0 : bin;
            float grad = sqrtf(dx * dx + dy * dy);
            hist[32 * (x - 1) + bin] += grad * gauss[x] * gauss[y];
        }
    }
    __syncthreads();

    // Combine histogram values
    for (int offset = 8; offset > 0; offset >>= 1) {
        for (int y = 0; y < 5; y++)
            hist[y * 32 + tx] += hist[(y + offset) * 32 + tx];
        __syncthreads();
    }
    
    // Recalculate the histogram for smoothness
    float *h = hist + 32;
    if (tx == 0)
        h[0] = 6 * hist[0] + 4 * (hist[1] + hist[31]) + (hist[2] + hist[30]);
    else if (tx == 31)
        h[tx] = 6 * hist[31] + 4 * (hist[0] + hist[30]) + (hist[1] + hist[29]);
    else
        h[tx] = 6 * hist[tx] + 4 * (hist[tx + 1] + hist[tx - 1]) + (hist[tx + 2] + hist[tx - 2]);
    __syncthreads();

    // Detect peaks in the histogram
    float v = h[tx];
    hist[tx] = (v > h[(tx + 1) & 31] && v >= h[(tx + 31) & 31]) ? v : 0.0f;
    __syncthreads();

    // Write back the key orientations 
    if (tx == 0) {
        float maxval1 = 0.0f, maxval2 = 0.0f;
        int i1 = -1, i2 = -1;
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }
        float val1 = h[(i1 + 1) & 31];
        float val2 = h[(i1 + 31) & 31];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Sift[bx + 5 * maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        if (maxval2 < 0.8f * maxval1)
            i2 = -1;
        if (i2 >= 0) {
            val1 = h[(i2 + 1) & 31];
            val2 = h[(i2 + 31) & 31];
            peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Sift[bx + 6 * maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        } else
            d_Sift[bx + 6 * maxPts] = i2;
    }
}