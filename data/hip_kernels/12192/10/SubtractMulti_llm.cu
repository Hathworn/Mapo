#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Calculate indices for x and y positions
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int sz = height * pitch;

    // Consolidate computation of linear index
    int p = y * pitch + x;
    
    // Check bounds before computation
    if (x < width && y < height)
    {
        // Simplifying and ensuring work per thread is independent
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
}