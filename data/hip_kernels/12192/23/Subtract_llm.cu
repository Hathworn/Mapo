#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Calculate global thread index without using deprecated __mul24
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;

    // Ensure indexing is within array bounds
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }

    // Remove unnecessary __syncthreads() as it's not needed here
}