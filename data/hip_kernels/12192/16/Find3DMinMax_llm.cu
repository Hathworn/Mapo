#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int pitch, int height)
{
    // Data cache with optimized shared memory utilization
    __shared__ float data1[3 * (MINMAX_W + 2)];
    __shared__ float data2[3 * (MINMAX_W + 2)];
    __shared__ float data3[3 * (MINMAX_W + 2)];
    __shared__ float ymin1[MINMAX_W + 2];
    __shared__ float ymin2[MINMAX_W + 2];
    __shared__ float ymin3[MINMAX_W + 2];
    __shared__ float ymax1[MINMAX_W + 2];
    __shared__ float ymax2[MINMAX_W + 2];
    __shared__ float ymax3[MINMAX_W + 2];

    // Current tile and apron limits, relative to row start
    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_W;
    const int xEnd = xStart + MINMAX_W - 1;
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;
    const int xEndClamped = min(xEnd, width - 1);
    const int memWid = MINMAX_W + 2;

    int memPos0 = (tx - WARP_SIZE + 1);
    int memPos1 = (tx - WARP_SIZE + 1);
    int yq = 0;
    unsigned int output = 0;

    for (int y = 0; y < 32 + 2; y++) {
        output >>= 1;
        const int memPos = yq * memWid + (tx - WARP_SIZE + 1);
        int yp = 32 * blockIdx.y + y - 1;
        yp = clamp(yp, 0, height - 1);
        int readStart = yp * pitch;

        // Load data into shared memory with boundary checks
        if (tx >= (WARP_SIZE - 1)) {
            if (xReadPos >= 0 && xReadPos < width) {
                data1[memPos] = d_Data1[readStart + xReadPos];
                data2[memPos] = d_Data2[readStart + xReadPos];
                data3[memPos] = d_Data3[readStart + xReadPos];
            } else {
                data1[memPos] = 0;
                data2[memPos] = 0;
                data3[memPos] = 0;
            }
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;
        if (y > 1) {
            if (tx < memWid) {
                // Calculate min/max using the shared memory data
                float min1 = fminf(fminf(data1[memPos0], data1[memPos1]), data1[memPos2]);
                float min2 = fminf(fminf(data2[memPos0], data2[memPos1]), data2[memPos2]);
                float min3 = fminf(fminf(data3[memPos0], data3[memPos1]), data3[memPos2]);
                float max1 = fmaxf(fmaxf(data1[memPos0], data1[memPos1]), data1[memPos2]);
                float max2 = fmaxf(fmaxf(data2[memPos0], data2[memPos1]), data2[memPos2]);
                float max3 = fmaxf(fmaxf(data3[memPos0], data3[memPos1]), data3[memPos2]);
                ymin1[tx] = min1;
                ymin2[tx] = fminf(min1, fminf(min2, min3));
                ymin3[tx] = min3;
                ymax1[tx] = max1;
                ymax2[tx] = fmaxf(max1, fmaxf(max2, max3));
                ymax3[tx] = max3;
            }
        }
        __syncthreads();

        if (y > 1) {
            if (tx < MINMAX_W) {
                if (xWritePos <= xEndClamped) {
                    // Compute final min/max and apply threshold
                    float minv = fminf(ymin2[tx], ymin2[tx + 2]);
                    minv = fminf(minv, fminf(ymin1[tx + 1], ymin3[tx + 1]));
                    minv = fminf(minv, fminf(data2[memPos0 + 1], data2[memPos2 + 1]));
                    minv = fminf(minv, d_Threshold[1]);
                    
                    float maxv = fmaxf(ymax2[tx], ymax2[tx + 2]);
                    maxv = fmaxf(maxv, fmaxf(ymax1[tx + 1], ymax3[tx + 1]));
                    maxv = fmaxf(maxv, fmaxf(data2[memPos0 + 1], data2[memPos2 + 1]));
                    maxv = fmaxf(maxv, d_Threshold[0]);
                    
                    if (data2[memPos1 + 1] < minv || data2[memPos1 + 1] > maxv) {
                        output |= 0x80000000;
                    }
                }
            }
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }
    
    if (tx < MINMAX_W && xWritePos < width) {
        int writeStart = blockIdx.y * pitch + xWritePos;
        d_Result[writeStart] = output;
    }
}