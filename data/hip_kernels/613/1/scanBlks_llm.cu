#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scanBlks(unsigned int *in, unsigned int *out, unsigned int n, unsigned int *blkSums)
{
    extern __shared__ int blkData[];
    int i1 = blockIdx.x * 2 * blockDim.x + threadIdx.x;
    int i2 = i1 + blockDim.x;

    // Load data to shared memory and zero-pad if necessary
    blkData[threadIdx.x] = (i1 < n) ? in[i1] : 0;
    blkData[threadIdx.x + blockDim.x] = (i2 < n) ? in[i2] : 0;
    __syncthreads();

    // Up-sweep phase
    for (int stride = 1; stride < 2 * blockDim.x; stride *= 2)
    {
        int blkDataIdx = (threadIdx.x + 1) * 2 * stride - 1;
        if (blkDataIdx < 2 * blockDim.x)
            blkData[blkDataIdx] += blkData[blkDataIdx - stride];
        __syncthreads();
    }

    // Down-sweep phase
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        int blkDataIdx = (threadIdx.x + 1) * 2 * stride - 1 + stride;
        if (blkDataIdx < 2 * blockDim.x)
            blkData[blkDataIdx] += blkData[blkDataIdx - stride];
        __syncthreads();
    }

    // Write the results to the output array
    if (i1 < n) out[i1] = blkData[threadIdx.x];
    if (i2 < n) out[i2] = blkData[threadIdx.x + blockDim.x];

    // Store block sum to blkSums
    if (blkSums != NULL && threadIdx.x == 0)
        blkSums[blockIdx.x] = blkData[2 * blockDim.x - 1];
}