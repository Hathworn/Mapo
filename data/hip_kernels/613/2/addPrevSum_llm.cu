#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel to add previous block sum to current block scan results
__global__ void addPrevSum(unsigned int* blkSumsScan, unsigned int* blkScans, unsigned int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n && threadIdx.x >= blockDim.x)
    {
        // Add the prefix sum of previous blocks to current block scans
        blkScans[i] += blkSumsScan[blockIdx.x - 1];
    }
}