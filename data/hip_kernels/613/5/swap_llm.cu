#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void swap(unsigned int *in, unsigned int *in_pos, unsigned int *out, unsigned int *out_pos, unsigned int n)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        // Use shared memory for faster access
        __shared__ unsigned int temp_storage1[1024];
        __shared__ unsigned int temp_storage2[1024];

        temp_storage1[threadIdx.x] = in[i];
        temp_storage2[threadIdx.x] = in_pos[i];

        __syncthreads();  // Ensure all threads have written to shared memory

        in[i] = out[i];
        in_pos[i] = out_pos[i];

        __syncthreads();  // Ensure before fetching from shared memory

        out[i] = temp_storage1[threadIdx.x];
        out_pos[i] = temp_storage2[threadIdx.x];
    }
}