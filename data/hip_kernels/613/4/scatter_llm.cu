#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Udacity HW 4
// Radix Sorting

__global__ void scatter(unsigned int *in, unsigned int *in_pos, unsigned int *out, unsigned int *out_pos, unsigned int n, unsigned int *d_histScan, unsigned int mask, unsigned int current_bits, unsigned int nBins)
{
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < n)
    {
        // Calculate the bin for each thread's element
        unsigned int bin = (in[tid] >> current_bits) & mask;
        unsigned int index = atomicAdd(&d_histScan[bin * gridDim.x + blockIdx.x], 1);
        
        // Write to the output array at the correct bin position
        out[index] = in[tid];
        out_pos[index] = in_pos[tid];
    }
}