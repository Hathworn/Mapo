#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Udacity HW 4
// Radix Sorting

__global__ void histogram(unsigned int* in, unsigned int* hist, int n, unsigned int nBins, unsigned int mask, unsigned int current_bits)
{
    extern __shared__ unsigned int s_local_hist[];

    // Initialize shared memory for local histogram
    for(int j = threadIdx.x; j < nBins; j += blockDim.x)
        s_local_hist[j] = 0;

    __syncthreads();

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        unsigned int bin = (in[i] >> current_bits) & mask;
        atomicAdd(&s_local_hist[bin], 1);
    }
    __syncthreads();

    // Copy the shared memory histogram to the global memory
    // Using coalesced access pattern for improved performance
    for (int bin = threadIdx.x; bin < nBins; bin += blockDim.x)
    {
        hist[bin * gridDim.x + blockIdx.x] = s_local_hist[bin];
    }
}