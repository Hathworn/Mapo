#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Udacity HW 4
// Radix Sorting

__global__ void exclusive_scan(unsigned int *in, unsigned int *out, int n)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n)
    {
        // Simplify subtraction by leveraging shared memory for in-place computation
        out[i] = (i > 0) ? in[i - 1] : 0; 
    }
}