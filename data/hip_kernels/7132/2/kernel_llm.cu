#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(float* data, size_t from, size_t to, size_t min, size_t max, size_t NX)
{
    size_t i = min + blockIdx.x * blockDim.x + threadIdx.x;
    size_t grid_stride = gridDim.x * blockDim.x; // Use grid stride for efficient looping

    while (i < max) {
        size_t row = i / NX; // Precompute row index
        size_t col = i % NX; // Precompute column index

        // Avoid modulus check with simple comparison
        if (col != 0 && col != NX - 1) {
            data[to + i] = 0.2 * (
                data[from + i]
                + data[from + i - 1]
                + data[from + i + 1]
                + data[from + i - NX]
                + data[from + i + NX]);
        }
        i += grid_stride; // Move to the next position with grid stride
    }
}