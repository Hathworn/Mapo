#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateDisplacements_k(float4 *Ui_t, float4 *Ui_tminusdt, float *M, float4 *Ri, float4 *Fi, int maxNumForces, float4 *ABC, unsigned int numPoints)
{
    int me_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (me_idx >= numPoints)
        return;

    float4 F = make_float4(0, 0, 0, 0);

    // Optimize force accumulation using registers
    #pragma unroll
    for (int i = 0; i < maxNumForces; i++)
    {
        float4 force_to_add = Fi[me_idx * maxNumForces + i];
        F.x += force_to_add.x;
        F.y += force_to_add.y;
        F.z += force_to_add.z;
    }

    // Minimize memory reads by utilizing registers
    float4 ABCi = ABC[me_idx];
    float4 Uit = Ui_t[me_idx];
    float4 Uitminusdt = Ui_tminusdt[me_idx];
    float4 R = Ri[me_idx];

    // Optimize calculations to a single line per component
    Ui_tminusdt[me_idx] = make_float4(
        ABCi.x * (R.x - F.x) + ABCi.y * Uit.x + ABCi.z * Uitminusdt.x,
        ABCi.x * (R.y - F.y) + ABCi.y * Uit.y + ABCi.z * Uitminusdt.y,
        ABCi.x * (R.z - F.z) + ABCi.y * Uit.z + ABCi.z * Uitminusdt.z,
        0
    );
}