#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void precalculateABC(float4* ABCm, float* M, float timestep, float alpha, unsigned int numPoints)
{
    int me_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (me_idx >= numPoints)
        return;

    float twodelta = timestep * 2.0f;
    float deltasqr = timestep * timestep;

    // Use registers to minimize repeated global memory access
    float Mii = M[me_idx];
    float Dii = alpha * Mii; 

    float invDelta = 1.0f / deltasqr;  // Pre-calculate to reduce division operations
    float invTwodelta = 1.0f / twodelta;

    float Ai = 1.0f / (Dii * invTwodelta + Mii * invDelta);
    float Bi = 2.0f * Mii * invDelta * Ai;
    float Ci = (Dii * invTwodelta) * Ai - 0.5f * Bi;

    // Directly store the result to global memory
    ABCm[me_idx] = make_float4(Ai, Bi, Ci, Mii);
}