#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < N) // Ensure within bounds
    {
        c[i] = a[i] + b[i]; // Perform addition
    }
}