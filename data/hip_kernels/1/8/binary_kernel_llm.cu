#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32


__global__ void binary_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    if (x < n_a && y < n_b) {
        __shared__ float temp[2 * NUM_THREADS];

        // Initialize shared memory
        temp[threadIdx.x] = 0.0f;
        temp[threadIdx.x + NUM_THREADS] = 0.0f;

        // Accumulate results
        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            int a = vg_a[x * pitch_a + offset] != 0.0f;
            int b = vg_b[y * pitch_b + offset] != 0.0f;
            if (a ^ b) {
                temp[threadIdx.x] += 1.0f;
            }
            if (a || b) {
                temp[threadIdx.x + NUM_THREADS] += 1.0f;
            }
        }

        __syncthreads();

        // Perform reduction
        for (size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
                temp[threadIdx.x + NUM_THREADS] += temp[threadIdx.x + stride + NUM_THREADS];
            }
            __syncthreads();
        }

        // Write result to global memory
        if (threadIdx.x == 0) {
            float val = temp[0];
            if (temp[NUM_THREADS] != 0.0f) {
                val /= temp[NUM_THREADS];
            }
            d[y * pitch_d + x] = val;
        }
    }
}