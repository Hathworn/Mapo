#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuSDNoTest(const float *vectsA, size_t na, const float *vectsB, size_t nb, size_t dim, const float *means, const float *numPairs, float *sds)
{
    size_t tx = threadIdx.x;
    size_t bx = blockIdx.x, by = blockIdx.y;
    float termA, termB;
    __shared__ float meanA, meanB, n;
    __shared__ float threadSumsA[NUMTHREADS], threadSumsB[NUMTHREADS];

    // Ensure block is within bounds
    if (bx >= na || by >= nb)
        return;

    // Load shared memory data
    if (tx == 0) {
        meanA = means[bx * nb * 2 + by * 2];
        meanB = means[bx * nb * 2 + by * 2 + 1];
        n = numPairs[bx * nb + by];
    }
    __syncthreads();

    // Initialize shared thread sums in parallel
    threadSumsA[tx] = 0.f;
    threadSumsB[tx] = 0.f;

    // Calculate thread-specific sums
    for (size_t offset = tx; offset < dim; offset += NUMTHREADS) {
        float a = vectsA[bx * dim + offset];
        float b = vectsB[by * dim + offset];

        termA = a - meanA;
        termB = b - meanB;
        threadSumsA[tx] += termA * termA;
        threadSumsB[tx] += termB * termB;
    }
    __syncthreads();

    // Reduce sums within block using warp-efficient approach
    for (size_t stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
        if (tx < stride) {
            threadSumsA[tx] += threadSumsA[tx + stride];
            threadSumsB[tx] += threadSumsB[tx + stride];
        }
        __syncthreads();
    }

    // Compute final results
    if (tx == 0) {
        sds[bx * nb * 2 + by * 2] = sqrtf(threadSumsA[0] / (n - 1.f));
        sds[bx * nb * 2 + by * 2 + 1] = sqrtf(threadSumsB[0] / (n - 1.f));
    }
}