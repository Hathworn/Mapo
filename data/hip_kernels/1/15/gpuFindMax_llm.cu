#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512

__global__ void gpuFindMax(int n, float *data, int threadWorkLoad, int *maxIndex)
{
    int j, k;
    int start = threadWorkLoad * threadIdx.x;
    int end = min(start + threadWorkLoad, n); // Reduce redundant checks
    __shared__ int maxIndices[NTHREADS];

    maxIndices[threadIdx.x] = -1;

    if(start >= n)
        return;

    int localMaxIndex = start;
    for (int i = start + 1; i < end; i++) {
        if (data[i] > data[localMaxIndex])
            localMaxIndex = i;
    }
    maxIndices[threadIdx.x] = localMaxIndex;
    __syncthreads();

    // Optimize reduction with unrolling
    for (int i = blockDim.x >> 1; i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            j = maxIndices[threadIdx.x];
            k = maxIndices[i + threadIdx.x];
            if ((j != -1) && (k != -1) && (data[j] < data[k]))
                maxIndices[threadIdx.x] = k;
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        *maxIndex = maxIndices[0];
    }
}