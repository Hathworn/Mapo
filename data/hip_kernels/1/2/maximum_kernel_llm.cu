#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

// Optimized kernel function using warp shuffle reduction
__global__ void maximum_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // Check bounds
    if (x < n_a && y < n_b) {
        __shared__ float temp[NUM_THREADS];

        temp[threadIdx.x] = 0.0;
        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float t = abs(vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset]);
            temp[threadIdx.x] = max(temp[threadIdx.x], t);
        }
        
        // Sync threads
        __syncthreads();
        
        // Using warp shuffle to reduce
        float max_val = temp[threadIdx.x];
        for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
            max_val = max(max_val, __shfl_down_sync(0xffffffff, max_val, stride));
        }

        // Write maximum value to global memory
        if (threadIdx.x == 0) {
            d[y * pitch_d + x] = max_val;
        }
    }
}