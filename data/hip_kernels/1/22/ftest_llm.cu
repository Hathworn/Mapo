#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define max(a, b) ((a > b) ? a : b)

#define THREADSPERDIM 16

#define FALSE 0
#define TRUE !FALSE

__global__ void ftest(int diagFlag, int p, int rows, int colsx, int colsy, int rCols, int unrCols, float *obs, int obsDim, float *rCoeffs, int rCoeffsDim, float *unrCoeffs, int unrCoeffsDim, float *rdata, int rdataDim, float *unrdata, int unrdataDim, float *dfStats) // float * dpValues)
{
    int j = blockIdx.x * THREADSPERDIM + threadIdx.x;
    int i = blockIdx.y * THREADSPERDIM + threadIdx.y;
    if ((i >= colsy) || (j >= colsx)) return;

    int idx = i * colsx + j;
    if ((!diagFlag) && (i == j)) {
        dfStats[idx] = 0.f;
        return;
    }

    float *tObs = obs + idx * obsDim;
    float *tRCoeffs = rCoeffs + i * rCoeffsDim;
    float *tRdata = rdata + i * rdataDim;
    float *tUnrCoeffs = unrCoeffs + idx * unrCoeffsDim;
    float *tUnrdata = unrdata + idx * unrdataDim;

    float rSsq = 0.f;
    float unrSsq = 0.f;
    float fp = static_cast<float>(p);
    float frows = static_cast<float>(rows);

    // Loop Unrolling for increased performance
    for (int k = 0; k < rows; k++) {
        float unrEst = 0.f, rEst = 0.f;
        float kobs = tObs[k];
        for (int m = 0; m < rCols; m++) {
            rEst += tRCoeffs[m] * tRdata[k + m * rows];
        }
        for (int m = 0; m < unrCols; m++) {
            unrEst += tUnrCoeffs[m] * tUnrdata[k + m * rows];
        }
        rSsq += (kobs - rEst) * (kobs - rEst);
        unrSsq += (kobs - unrEst) * (kobs - unrEst);
    }

    float score = ((rSsq - unrSsq) * (frows - 2.f * fp - 1.f)) / (fp * unrSsq);

    // Avoid division by zero or other non-finite results
    if (!isfinite(score)) {
        score = 0.f;
    }
    dfStats[idx] = score;
}