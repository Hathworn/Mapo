#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuSD(const float * vectsA, size_t na, const float * vectsB, size_t nb, size_t dim, const float * means, const float * numPairs, float * sds)
{
    size_t tx = threadIdx.x;
    size_t bx = blockIdx.x, by = blockIdx.y;

    // Early return if block indices exceed available data
    if((bx >= na) || (by >= nb)) return;

    __shared__ float meanA, meanB, n;
    __shared__ float threadSumsA[NUMTHREADS], threadSumsB[NUMTHREADS];

    if(tx == 0) {
        meanA = means[bx*nb*2+by*2];
        meanB = means[bx*nb*2+by*2+1];
        n = numPairs[bx*nb+by];
    }
    __syncthreads();

    // Initialize thread sums
    float sumA = 0.f, sumB = 0.f;

    // Unroll the loop for performance optimization
    for(size_t offset = tx; offset < dim; offset += NUMTHREADS) {
        float a = vectsA[bx * dim + offset];
        float b = vectsB[by * dim + offset];
        if(!(isnan(a) || isnan(b))) {
            float termA = a - meanA;
            float termB = b - meanB;
            sumA += termA * termA;
            sumB += termB * termB;
        }
    }
    threadSumsA[tx] = sumA;
    threadSumsB[tx] = sumB;
    __syncthreads();

    // Reduce within shared memory using binary tree reduction
    for(size_t stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
        if(tx < stride) {
            threadSumsA[tx] += threadSumsA[tx + stride];
            threadSumsB[tx] += threadSumsB[tx + stride];
        }
        __syncthreads();
    }

    // Only thread 0 updates the result
    if(tx == 0) {
        sds[bx*nb*2+by*2]   = sqrtf(threadSumsA[0] / (n - 1.f));
        sds[bx*nb*2+by*2+1] = sqrtf(threadSumsB[0] / (n - 1.f));
    }
}