#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void get_mi(int nbins, int nsamples, int nx, float * x_bin_scores, int pitch_x_bin_scores, float * entropies_x, int ny, float * y_bin_scores, int pitch_y_bin_scores, float * entropies_y, float * mis, int pitch_mis)
{
    int col_x = blockDim.x * blockIdx.x + threadIdx.x;
    int col_y = blockDim.y * blockIdx.y + threadIdx.y;
    
    if((col_x >= nx) || (col_y >= ny))
        return;

    float mi = 0.f;
    float *x_bins = x_bin_scores + col_x * pitch_x_bin_scores;
    float *y_bins = y_bin_scores + col_y * pitch_y_bin_scores;

    // Unrolling loop for nbins
    for(int i = 0; i < nbins; i++) {
        #pragma unroll
        for(int j = 0; j < nbins; j++) {
            float prob = 0.f;
            for(int k = 0; k < nsamples; k++)
                prob += x_bins[k * nbins + i] * y_bins[k * nbins + j];
            prob /= (float)nsamples;

            float logp = (prob <= 0.f) ? 0.f : __log2f(prob);
            mi += prob * logp;
        }
    }

    mi += entropies_x[col_x] + entropies_y[col_y];
    (mis + col_y * pitch_mis)[col_x] = mi;
}