#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuPMCCNoTest(const float * vectsa, size_t na, const float * vectsb, size_t nb, size_t dim, const float * numPairs, const float * means, const float * sds, float * correlations)
{
    size_t x = blockIdx.x, y = blockIdx.y, tx = threadIdx.x;
    float a, b, n, scoreA, scoreB;
    __shared__ float meanA, meanB, sdA, sdB;
    __shared__ float threadSums[NUMTHREADS];

    if((x >= na) || (y >= nb))
        return;

    if(tx == 0) {
        meanA = means[x * nb * 2 + y * 2];
        meanB = means[x * nb * 2 + y * 2 + 1];
        sdA = sds[x * nb * 2 + y * 2];
        sdB = sds[x * nb * 2 + y * 2 + 1];
        n = numPairs[x * nb + y];
    }
    __syncthreads();

    float localSum = 0.f;
    for(size_t offset = tx; offset < dim; offset += NUMTHREADS) {
        a = vectsa[x * dim + offset];
        b = vectsb[y * dim + offset];

        scoreA = (a - meanA) / sdA;
        scoreB = (b - meanB) / sdB;
        localSum += scoreA * scoreB;
    }
    
    threadSums[tx] = localSum;
    __syncthreads();

    for(size_t stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
        if(tx < stride) {
            threadSums[tx] += threadSums[tx + stride];
        }
        __syncthreads();
    }
    
    if(tx == 0) {
        correlations[x * nb + y] = threadSums[0] / (n - 1.f);
    }
}