#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void euclidean_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x, y = blockIdx.y;

    // Initialize distance for diagonal elements
    if ((x == y) && (x < n_a) && (threadIdx.x == 0))
        d[y * pitch_d + x] = 0.0;

    // Compute element distances only for x < y
    if (y < n_a && x < y) {
        __shared__ float temp[NUM_THREADS];

        float sum = 0.0;
        // Calculate partial Euclidean distance for each thread
        for (size_t offset = threadIdx.x; offset < k; offset += NUM_THREADS) {
            float t = vg_a[x * pitch_a + offset] - vg_a[y * pitch_a + offset];
            sum += (t * t);
        }
        temp[threadIdx.x] = sum;

        __syncthreads();

        // Perform reduction in shared memory
        for (size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride)
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            __syncthreads();
        }

        // Write the result to global memory
        if (threadIdx.x == 0) {
            float s = sqrt(temp[0]);
            d[y * pitch_d + x] = s;
            d[x * pitch_d + y] = s;
        }
    }
}