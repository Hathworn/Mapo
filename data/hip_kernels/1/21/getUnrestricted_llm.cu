#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define max(a, b) ((a > b)?a:b)

#define THREADSPERDIM   16

#define FALSE 0
#define TRUE !FALSE

// Kernel function optimized for performance
__global__ void getUnrestricted(int countx, int county, int rows, int cols, float * mX, int mXdim, float * vY, int vYdim, float * mQ, int mQdim, float * mR, int mRdim, float * vectB, int vectBdim) {
    
    int n = blockIdx.x * THREADSPERDIM + threadIdx.x;
    int m = blockIdx.y * THREADSPERDIM + threadIdx.y;
    if((m >= county) || (n >= countx)) return;

    float * X = mX + (m * countx + n) * mXdim;
    float * Y = vY + (m * countx + n) * vYdim;
    float * B = vectB + (m * countx + n) * vectBdim;
    float * Q = mQ + (m * countx + n) * mQdim;
    float * R = mR + (m * countx + n) * mRdim;
    
    // Initialize intercepts and Q
    for(int i = 0; i < rows; i++) {
        X[i] = 1.f;
        for(int j = 0; j < cols; j++) 
            Q[i + j * rows] = X[i + j * rows];
    }

    float *colj, *coli, sum, invnorm;
    
    // Gram-Schmidt process to find Q
    for(int j = 0; j < cols; j++) {
        colj = Q + rows * j;
        
        for(int i = 0; i < j; i++) {
            coli = Q + rows * i;
            sum = 0.f;
            for(int k = 0; k < rows; k++)
                sum += coli[k] * colj[k];
            for(int k = 0; k < rows; k++)
                colj[k] -= sum * coli[k];
        }
        
        sum = 0.f;
        for(int i = 0; i < rows; i++)
            sum += colj[i] * colj[i];
        invnorm = rsqrtf(sum); // Utilize rsqrt for optimized inversion
        for(int i = 0; i < rows; i++)
            colj[i] *= invnorm;
    }

    // Back substitution to solve Rx = B
    for(int i = cols - 1; i > -1; i--) {
        float * colQ = Q + i * rows;
        
        // Compute R = Q^t * X
        for(int j = 0; j < cols; j++) {
            float * colX = X + j * rows;
            sum = 0.f;
            for(int k = 0; k < rows; k++)
                sum += colQ[k] * colX[k];
            R[i + j * cols] = sum;
        }
        
        sum = 0.f;
        // Compute Q^t * Y -> B
        for(int j = 0; j < rows; j++)
            sum += colQ[j] * Y[j];
        
        // Back substitution to find the solution
        for(int j = cols - 1; j > i; j--)
            sum -= R[i + j * cols] * B[j];
        
        B[i] = sum / R[i + i * cols];
    }
}