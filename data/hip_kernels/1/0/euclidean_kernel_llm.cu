#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 16

__global__ void euclidean_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // Check if within bounds
    if (x < n_a && y < n_b) {
        __shared__ float temp[NUM_THREADS];
        float sum = 0.0f; // Use register to accumulate

        // Load and accumulate in registers
        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float t = vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset];
            sum += (t * t);
        }

        // Store accumulated result in shared memory
        temp[threadIdx.x] = sum;
        __syncthreads();

        // Reduce using shared memory
        for (size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Write the final result to global memory
        if (threadIdx.x == 0) {
            d[y * pitch_d + x] = sqrt(temp[0]);
        }
    }
}