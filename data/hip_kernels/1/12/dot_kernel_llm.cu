#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void dot_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p) 
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // If all elements need to be computed
    if (x < n_a && y < n_b) {
        __shared__ float temp[NUM_THREADS];
        float sum = 0.0f;  // Use register for partial sum

        // Unroll inner loop for efficiency
        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            sum += vg_a[x * pitch_a + offset] * vg_b[y * pitch_b + offset];
        }

        temp[threadIdx.x] = sum;
        __syncthreads();

        // Reduce using shared memory
        for (size_t stride = NUM_THREADS >> 1; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Write result to global memory
        if (threadIdx.x == 0) {
            d[y * pitch_d + x] = temp[0];
        }
    }
}