#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float do_fraction(float numer, float denom) {
    float result = 0.f;
    if ((numer == denom) && (numer != 0.f))
        result = 1.f;
    else if (denom != 0.f)
        result = numer / denom;
    return result;
}

__global__ void get_bin_scores(int nbins, int order, int nknots, float *knots, int nsamples, int nx, float *x, int pitch_x, float *bins, int pitch_bins) {
    int col_x = blockDim.x * blockIdx.x + threadIdx.x;
    if (col_x >= nx)
        return;

    float *in_col = x + col_x * pitch_x;
    float *bin_col = bins + col_x * pitch_bins;

    for (int k = 0; k < nsamples; k++, bin_col += nbins) {
        float z = in_col[k];
        int i0 = (int)floorf(z) + order - 1;
        if (i0 >= nbins)
            i0 = nbins - 1;

        bin_col[i0] = 1.f;

        // Use shared memory to reduce global memory accesses
        extern __shared__ float shared_bins[];
        shared_bins[threadIdx.x] = 0.0f;

        for (int i = 2; i <= order; i++) {
            for (int j = i0 - i + 1; j <= i0; j++) {
                float rd = do_fraction(knots[j + i] - z, knots[j + i] - knots[j + 1]);

                float term1 = 0.f;
                if ((j >= 0) && (j < nbins) && (j < nknots) && (j + i - 1 >= 0) && (j < nknots)) {
                    float ld = do_fraction(z - knots[j], knots[j + i - 1] - knots[j]);
                    term1 = ld * bin_col[j];
                }

                float term2 = 0.f;
                if ((j + 1 >= 0) && (j + 1 < nbins) && (j + 1 < nknots) && (j + i >= 0) && (j + i < nknots)) {
                    term2 = rd * bin_col[j + 1];
                }
                bin_col[j] = term1 + term2;
                
                // Cache result in shared memory
                if (j < blockDim.x)
                    shared_bins[j] = bin_col[j];
            }

            // Synchronize threads to ensure shared memory is updated
            __syncthreads();

            // Copy back from shared memory to global memory
            for (int j = i0 - i + 1; j <= i0; j++) {
                if (j < blockDim.x)
                    bin_col[j] = shared_bins[j];
            }

            // Synchronize again to prevent data racing
            __syncthreads();
        }
    }
}