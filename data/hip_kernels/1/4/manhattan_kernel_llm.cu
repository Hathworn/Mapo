#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void manhattan_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // If all element is to be computed
    if (x < n_a && y < n_b) {
        __shared__ float temp[NUM_THREADS];
        
        // Initialize shared memory
        temp[threadIdx.x] = 0.0;

        // Calculate manhattan distance using coalesced access
        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float t = abs(vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset]);
            temp[threadIdx.x] += t;
        }

        // Sync with other threads
        __syncthreads();

        // Reduce using loop unrolling for better performance
        for (size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Write to global memory
        if (threadIdx.x == 0) {
            d[y * pitch_d + x] = temp[0];
        }
    }
}