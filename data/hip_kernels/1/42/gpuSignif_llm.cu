#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuSignif(const float *gpuNumPairs, const float *gpuCorrelations, size_t n, float *gpuTScores) {
    size_t bx = blockIdx.x, tx = threadIdx.x;
    int idx = bx * NUMTHREADS * THREADWORK + tx * THREADWORK;

    // Use shared memory to reduce global memory access latency
    __shared__ float sharedNumPairs[THREADWORK];
    __shared__ float sharedCorrelations[THREADWORK];

    // Load data into shared memory
    for (int i = 0; i < THREADWORK; i++) {
        if (idx + i < n) {
            sharedNumPairs[i] = gpuNumPairs[idx + i];
            sharedCorrelations[i] = gpuCorrelations[idx + i];
        }
    }
    __syncthreads(); // Ensure all threads have loaded the data

    for (int i = 0; i < THREADWORK; i++) {
        if (idx + i >= n)
            break;

        float npairs = sharedNumPairs[i];
        float cor = sharedCorrelations[i];
        float radicand = (npairs - 2.f) / (1.f - cor * cor);
        gpuTScores[idx + i] = cor * sqrtf(radicand);
    }
}