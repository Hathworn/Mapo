#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void noNAsPmccMeans(int nRows, int nCols, float * a, float * means)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if(col >= nCols) return;

    int inOffset = col * nRows;
    int outOffset = threadIdx.x * blockDim.y + threadIdx.y;
    
    extern __shared__ float threadSums[];
    float sum = 0.f;

    for(int i = threadIdx.y; i < nRows; i += blockDim.y) {
        sum += a[inOffset + i];
    }
    
    threadSums[outOffset] = sum;
    __syncthreads();

    if(threadIdx.y == 0) {
        float totalSum = 0.f;
        // Efficient reduction in shared memory
        for(int i = 0; i < blockDim.y; ++i) {
            totalSum += threadSums[outOffset + i];
        }
        means[col] = totalSum / (float)nRows;
    }
}