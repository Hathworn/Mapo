#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuMeansNoTest(const float *vectsA, size_t na, const float *vectsB, size_t nb, size_t dim, float *means, float *numPairs) {
    size_t bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x;
    float a, b;

    // Use shared memory to accumulate sums and counts
    __shared__ float threadSumsA[NUMTHREADS];
    __shared__ float threadSumsB[NUMTHREADS];
    __shared__ float count[NUMTHREADS];

    if ((bx >= na) || (by >= nb))
        return;

    threadSumsA[tx] = 0.f;
    threadSumsB[tx] = 0.f;
    count[tx] = 0.f;

    // Loop through dimensions in chunks of NUMTHREADS
    for (size_t offset = tx; offset < dim; offset += NUMTHREADS) {
        a = vectsA[bx * dim + offset];
        b = vectsB[by * dim + offset];

        // Accumulate sums
        threadSumsA[tx] += a;
        threadSumsB[tx] += b;
        // Count elements processed by this thread
        count[tx] += 1.f;
    }
    __syncthreads();

    // Reduce sums and counts across threads
    for (size_t stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
        if (tx < stride) {
            threadSumsA[tx] += threadSumsA[tx + stride];
            threadSumsB[tx] += threadSumsB[tx + stride];
            count[tx] += count[tx + stride];
        }
        __syncthreads();
    }

    // Use one thread to compute final results
    if (tx == 0) {
        means[bx * nb * 2 + by * 2] = threadSumsA[0] / count[0];
        means[bx * nb * 2 + by * 2 + 1] = threadSumsB[0] / count[0];
        numPairs[bx * nb + by] = count[0];
    }
}