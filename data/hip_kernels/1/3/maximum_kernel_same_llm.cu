#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void maximum_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x, y = blockIdx.y;

    // Set self-comparison result to 0.0
    if (x == y && x < n_a && threadIdx.x == 0) {
        d[y * pitch_d + x] = 0.0;
    }

    // Compute maximum distance for x < y
    if (y < n_a && x < y) {
        __shared__ float temp[NUM_THREADS];

        // Initialize shared memory
        temp[threadIdx.x] = 0.0;
        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float t = fabsf(vg_a[x * pitch_a + offset] - vg_a[y * pitch_a + offset]);
            temp[threadIdx.x] = max(t, temp[threadIdx.x]);
        }

        // Sync threads to ensure all have completed the above loop
        __syncthreads();

        // Reduction to find the maximum in the block
        for (size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                temp[threadIdx.x] = max(temp[threadIdx.x], temp[threadIdx.x + stride]);
            }
            __syncthreads();
        }

        // Write final result from shared memory to global memory
        if (threadIdx.x == 0) {
            float s = temp[0];
            d[y * pitch_d + x] = s;
            d[x * pitch_d + y] = s;
        }
    }
}