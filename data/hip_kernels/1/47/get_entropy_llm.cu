#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void get_entropy(int nbins, int nsamples, int nx, float *bin_scores, int pitch_bin_scores, float *entropies) {
    int col_x = blockDim.x * blockIdx.x + threadIdx.x;

    if (col_x >= nx) return;

    float *in_col = bin_scores + col_x * pitch_bin_scores;
    float entropy = 0.f;

    // Use shared memory for reduction
    extern __shared__ float shared_probs[];

    for (int i = 0; i < nbins; i++) {
        shared_probs[threadIdx.x] = 0.f;
        for (int j = threadIdx.x; j < nsamples; j += blockDim.x) {
            shared_probs[threadIdx.x] += in_col[j * nbins + i];
        }
        
        // Synchronize threads to ensure all contributions are summed
        __syncthreads();

        // Reduce within block
        for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
            if (threadIdx.x < offset) {
                shared_probs[threadIdx.x] += shared_probs[threadIdx.x + offset];
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            float prob = shared_probs[0] / nsamples;
            float logp = (prob <= 0.f) ? 0.f : __log2f(prob);
            entropy += prob * logp;
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        entropies[col_x] = -entropy;
    }
}