#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void dot_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // If all elements are to be computed
    if (y < n_a && x <= y) {
        __shared__ float temp[NUM_THREADS];

        float sum = 0.0f;
        // Unroll loop for efficiency
        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            sum += vg_a[x * pitch_a + offset] * vg_a[y * pitch_a + offset];
        }
        temp[threadIdx.x] = sum;

        // Sync with other threads
        __syncthreads();

        // Reduce using warp-level primitives
        for (size_t stride = warpSize >> 1; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) temp[threadIdx.x] += temp[threadIdx.x + stride];
            __syncthreads();
        }

        // Write to global memory
        if (threadIdx.x == 0) {
            d[y * pitch_d + x] = temp[0];
            d[x * pitch_d + y] = temp[0];
        }
    }
}