#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void binary_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    if (x == y && x < n_a && threadIdx.x == 0) {
        d[y * pitch_d + x] = 0.0;
    }

    if (y < n_a && x < y) {
        // Use a single shared memory array to reduce memory usage
        __shared__ float temp[NUM_THREADS * 2];
        size_t tid = threadIdx.x;
        
        temp[tid] = 0.0f;
        temp[tid + NUM_THREADS] = 0.0f;

        // Unroll loop to improve performance
        for (size_t offset = tid; offset < k; offset += blockDim.x) {
            int a = vg_a[x * pitch_a + offset] != 0.0;
            int b = vg_a[y * pitch_a + offset] != 0.0;
            if (a ^ b) {
                temp[tid] += 1.0f;
            }
            if (a || b) {
                temp[tid + NUM_THREADS] += 1.0f;
            }
        }

        __syncthreads();

        // Optimize reduction with a single loop
        for (size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if (tid < stride) {
                temp[tid] += temp[tid + stride];
                temp[tid + NUM_THREADS] += temp[tid + stride + NUM_THREADS];
            }
            __syncthreads();
        }

        // Write results to global memory
        if (tid == 0) {
            float val = temp[0];
            if (temp[NUM_THREADS] != 0.0f) {
                val /= temp[NUM_THREADS];
            }
            d[y * pitch_d + x] = val;
            d[x * pitch_d + y] = val;
        }
    }
}