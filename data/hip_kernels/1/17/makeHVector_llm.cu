#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512

__global__ void makeHVector(int rows, float * input, float * output)
{
    int i;
    float elt, sum;
    __shared__ float beta, sums[NTHREADS];

    // Avoids unnecessary thread execution outside the needed range
    if(threadIdx.x >= rows)
        return;
    
    sum = 0.f;
    // Unroll loop partially for better performance
    for(i = threadIdx.x ; i < rows; i += NTHREADS) {
        if((threadIdx.x == 0) && (i == 0))
            continue;
        elt = input[i];
        output[i] = elt;
        sum += elt * elt;
    }
    sums[threadIdx.x] = sum;
    __syncthreads();

    // Optimize reduction with loop unrolling
    for(i = blockDim.x >> 1; i > 0 ; i >>= 1) {
        if(threadIdx.x < i) 
            sums[threadIdx.x] += sums[threadIdx.x + i];
        __syncthreads();
    }

    if(threadIdx.x == 0) {
        elt = input[0];
        float norm = sqrtf(elt * elt + sums[0]);

        // Use a single branch to update elt
        elt += (elt > 0) ? norm : -norm;

        output[0] = elt;

        norm = elt * elt + sums[0];
        beta = sqrtf(2.f / norm);
    }
    __syncthreads();

    // Vectorization opportunity by eliminating loop
    for(i = threadIdx.x; i < rows; i += NTHREADS)
        output[i] *= beta;
}