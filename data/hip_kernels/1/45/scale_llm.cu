#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale(float knot_max, int nx, int nsamples, float * x, int pitch_x)
{
    int col_idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (col_idx >= nx) return;

    float min = FLT_MAX, max = -FLT_MAX;
    float *col = x + col_idx * pitch_x;

    // Use cache-efficient access pattern and parallel reduction if possible
    for (int i = threadIdx.x; i < nsamples; i += blockDim.x) {
        float val = col[i];
        atomicMin(&min, val);
        atomicMax(&max, val);
    }
    __shared__ float s_min, s_max; // Shared memory for cache locality
    if (threadIdx.x == 0) {
        s_min = min;
        s_max = max;
    }
    __syncthreads();

    float delta = s_max - s_min;
    if (delta > 0.0f) {
        for (int i = threadIdx.x; i < nsamples; i += blockDim.x) {
            col[i] = (knot_max * (col[i] - s_min)) / delta;
        }
    }
}