#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512

__global__ void gpuSwapCol(int rows, float * dArray, int coli, int * dColj, int * dPivot)
{
    int rowIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for frequently accessed read-write variables (optimization)
    __shared__ int colj;
    __shared__ int iholder;

    if(threadIdx.x == 0) {
        colj = coli + (*dColj);
    }
    __syncthreads();

    if(rowIndex < rows) {  // Simplified condition
        float fholder = dArray[rowIndex + coli * rows];
        dArray[rowIndex + coli * rows] = dArray[rowIndex + colj * rows];
        dArray[rowIndex + colj * rows] = fholder;
    }

    // Synchronize here to ensure all threads are done before handling pivot
    __syncthreads();

    if(blockIdx.x == 0 && threadIdx.x == 0) {
        iholder = dPivot[coli];
        dPivot[coli] = dPivot[colj];
        dPivot[colj] = iholder;
    }
}