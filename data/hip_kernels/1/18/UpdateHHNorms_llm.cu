#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512

// Updates the column norms by subtracting the Hadamard-square of the
// Householder vector.
//
// N.B.:  Overflow incurred in computing the square should already have
// been detected in the original norm construction.

__global__ void UpdateHHNorms(int cols, float *dV, float *dNorms) {
    // Ensure each thread processes multiple elements for better efficiency
    int colIndex = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = colIndex; i < cols; i += stride) {
        float val = dV[i];
        dNorms[i] -= val * val;
    }
}