#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuKendall(const float * a, size_t na, const float * b, size_t nb, size_t sampleSize, double * results)
{
    size_t tx = threadIdx.x, ty = threadIdx.y;
    size_t bx = blockIdx.x, by = blockIdx.y;
    size_t rowa = bx * sampleSize, rowb = by * sampleSize;
    __shared__ float threadSums[NUMTHREADS * NUMTHREADS];

    float concordant = 0.f;

    // Perform computation in parallel
    for(size_t i = tx; i < sampleSize - 1; i += NUMTHREADS) {
        for(size_t j = i + 1 + ty; j < sampleSize; j += NUMTHREADS) {
            concordant += ((a[rowa + j] > a[rowa + i] && b[rowb + j] > b[rowb + i]) ||
                           (a[rowa + j] < a[rowa + i] && b[rowb + j] < b[rowb + i]) ||
                           (a[rowa + j] == a[rowa + i] && b[rowb + j] == b[rowb + i]));
        }
    }
    threadSums[ty * NUMTHREADS + tx] = concordant;

    __syncthreads();

    // Reduce using shared memory
    if (ty == 0) {
        for (size_t i = 1; i < NUMTHREADS; ++i) {
            threadSums[tx * NUMTHREADS] += threadSums[i * NUMTHREADS + tx];
        }
    }

    __syncthreads();

    if (tx == 0 && ty == 0) {
        concordant = threadSums[0];
        float denom = (float)sampleSize;
        denom = (denom * (denom - 1.f)) / 2.f;
        float discordant = denom - concordant;
        float numer = concordant - discordant;
        results[by * na + bx] = ((double)numer) / ((double)denom);
    }
}