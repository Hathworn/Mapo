#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__device__ int dIsSignificant(float signif, int df) {
    float tcutoffs[49] = {
        637.000, 31.600, 2.920, 8.610, 6.869, 5.959, 5.408, 5.041, 4.781,
        4.587, 4.437, 4.318, 4.221, 4.140, 4.073, 4.015, 3.965, 3.922,
        3.883, 3.850, 3.819, 3.792, 3.768, 3.745, 3.725, 3.707, 3.690,
        3.674, 3.659, 3.646, 3.622, 3.601, 3.582, 3.566, 3.551, 3.538,
        3.526, 3.515, 3.505, 3.496, 3.476, 3.460, 3.447, 3.435, 3.416,
        3.390, 3.357, 3.340, 3.290
    };

    size_t index = 0;
    if (df <= 0) return 0;
    else if (df <= 30) index = df - 1;
    else if (df <= 50) index = 30 + (df >> 1) - 16;
    else if (df <= 70) index = 40 + (df - 55) / 5;
    else if (df <= 80) index = 44;
    else if (df <= 100) index = 45;
    else if (df <= 150) index = 46;
    else if (df <= 200) index = 47;
    else index = 48;

    return fabsf(signif) >= tcutoffs[index];
}

__global__ void dUpdateSignif(const float * gpuData, size_t n, float * gpuResults) {
    size_t bx = blockIdx.x, tx = threadIdx.x;
    size_t start = bx * NUMTHREADS * THREADWORK + tx * THREADWORK;

    // Unroll the loop for better performance
    #pragma unroll
    for(int i = 0; i < THREADWORK; i++) {
        size_t index = start + i;
        if (index >= n) break;

        size_t inrow = index * 5;
        size_t outrow = index * 6;
        
        float cor = gpuData[inrow + 3];
        float npairs = gpuData[inrow + 4];
        float tscore = (cor >= 0.999) ? 10000.0f : cor * sqrtf((npairs - 2.f) / (1.f - cor * cor));

        if (dIsSignificant(tscore, (int)npairs)) {
            for (int j = 0; j < 6; ++j) {
                gpuResults[outrow + j] = gpuData[inrow + j];
            }
            gpuResults[outrow + 4] = tscore;
        } else {
            gpuResults[outrow] = -1.f;
        }
    }
}