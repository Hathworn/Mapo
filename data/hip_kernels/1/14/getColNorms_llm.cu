#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512

// Optimized kernel to update column norms by calculating the squared sum of the Householder vector.
__global__ void getColNorms(int rows, int cols, float *da, int lda, float *colNorms)
{
    int colIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (colIndex >= cols) return;

    float sum = 0.f;
    float *col = da + colIndex * lda;

    // Use unrolled loop for better performance
    for (int i = 0; i < rows - 3; i += 4) {
        sum += col[i] * col[i];
        sum += col[i + 1] * col[i + 1];
        sum += col[i + 2] * col[i + 2];
        sum += col[i + 3] * col[i + 3];
    }

    // Handle remaining elements
    for (int i = (rows / 4) * 4; i < rows; i++) {
        sum += col[i] * col[i];
    }

    colNorms[colIndex] = sum;
}