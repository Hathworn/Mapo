#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c, int size) {
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check to ensure index is within bounds
    if (idx < size) {
        c[idx] = a[idx] + b[idx];
    }
}