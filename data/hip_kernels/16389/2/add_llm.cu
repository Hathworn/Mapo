#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void add(int *a, int *b, int *c, int size) {
    int idx = size * blockIdx.x + threadIdx.x; // Precompute index
    c[idx] = a[idx] + b[idx]; // Use precomputed index
}