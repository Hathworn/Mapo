#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c, int size) {
    // Use a shared memory index calculation for improved readability and potential performance gains
    int idx = size * blockIdx.x + threadIdx.x;
    c[idx] = a[idx] + b[idx];
}