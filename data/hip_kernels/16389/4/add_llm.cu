#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    // Make sure to stay within bounds
    if (idx < gridDim.x * blockDim.x) {
        c[idx] = a[idx] + b[idx];
    }
}