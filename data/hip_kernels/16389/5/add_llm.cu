#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c, int size) {
    // Calculate global thread ID once to avoid redundant computation
    int idx = size * blockIdx.x + threadIdx.x;
    // Perform the addition operation
    c[idx] = a[idx] + b[idx];
}