#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void query_ball_point_gpu(int b, int n, int m, float radius, int nsample, const float *xyz1, const float *xyz2, int *idx, int *pts_cnt) {
    int batch_index = blockIdx.x;
    xyz1 += n*3*batch_index;
    xyz2 += m*3*batch_index;
    idx += m*nsample*batch_index;
    pts_cnt += m*batch_index;

    int index = threadIdx.x; 
    int stride = blockDim.x;
    
    __shared__ float s_xyz2[1024 * 3]; // Utilize shared memory for xyz2
    
    for (int j = index; j < m; j += stride) {
        int cnt = 0;
        if(threadIdx.x < m) {
            s_xyz2[threadIdx.x * 3 + 0] = xyz2[j * 3 + 0];
            s_xyz2[threadIdx.x * 3 + 1] = xyz2[j * 3 + 1];
            s_xyz2[threadIdx.x * 3 + 2] = xyz2[j * 3 + 2];
        }
        __syncthreads(); // Synchronize shared memory load
        
        float x2 = s_xyz2[threadIdx.x * 3 + 0];
        float y2 = s_xyz2[threadIdx.x * 3 + 1];
        float z2 = s_xyz2[threadIdx.x * 3 + 2];
        
        for (int k = 0; k < n; ++k) {
            if (cnt == nsample) break;
            float x1 = xyz1[k * 3 + 0];
            float y1 = xyz1[k * 3 + 1];
            float z1 = xyz1[k * 3 + 2];
            float dx = x2 - x1;
            float dy = y2 - y1;
            float dz = z2 - z1;
            float d2 = dx * dx + dy * dy + dz * dz;
            if (d2 < radius * radius) {
                if (cnt == 0) {
                    for (int l = 0; l < nsample; ++l)
                        idx[j * nsample + l] = k;
                }
                idx[j * nsample + cnt] = k;
                cnt += 1;
            }
        }
        pts_cnt[j] = cnt;
        __syncthreads(); // Ensure completion before next iteration
    }
}