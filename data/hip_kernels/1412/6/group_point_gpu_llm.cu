#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n * c * batch_index;
    idx += m * nsample * batch_index;
    out += m * nsample * c * batch_index;

    int j = blockIdx.y * blockDim.x + threadIdx.x; // Calculate the global thread position for `j`

    if (j < m) { // Ensure `j` is within bounds
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            for (int l = 0; l < c; ++l) {
                out[j * nsample * c + k * c + l] = points[ii * c + l];
            }
        }
    }
}