#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void maximumCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height) {
    // Calculate the global index for x and y efficiently
    const uint32_t x = threadIdx.x + blockIdx.x * blockDim.x;
    const uint32_t y = threadIdx.y + blockIdx.y * blockDim.y;

    // Perform bounds check to avoid out-of-bounds access
    if (x < width && y < height) {
        // Directly compute input and output indices
        const uint32_t indexIn1 = y * rowSizeIn1 + x;
        const uint32_t indexIn2 = y * rowSizeIn2 + x;
        const uint32_t indexOut = y * rowSizeOut + x;

        // Optimize the conditional assignment using in-place computation
        out[indexOut] = max(in1[indexIn1], in2[indexIn2]);
    }
}