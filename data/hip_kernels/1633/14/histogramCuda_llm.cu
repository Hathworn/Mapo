#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histogramCuda( const uint8_t * data, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * histogram )
{
    // Calculate global thread index
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Check for boundaries
    if ( x < width && y < height ) {
        // Calculate linear index
        const uint32_t id = y * rowSize + x;

        // Use shared memory for intermediate histogram results (assuming 256 bins)
        extern __shared__ uint32_t sharedHist[];
        uint32_t bin = data[id];
        atomicAdd(&sharedHist[bin], 1);

        // Ensure all threads have updated shared memory
        __syncthreads();

        // Use first thread in each block to update the global histogram
        if (threadIdx.x == 0 && threadIdx.y == 0) {
            for (int i = 0; i < 256; ++i) {
                atomicAdd(&histogram[i], sharedHist[i]);
            }
        }
    }
}