#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Optimize by ensuring in-bounds checks happen once and use shared memory (if possible, assume data fits in shared memory)
    if (x < width && y < height)
    {
        uint32_t indexIn = y * rowSizeIn + x;
        uint32_t indexOut = y * rowSizeOut + x;
        out[indexOut] = in[indexIn];
    }
}