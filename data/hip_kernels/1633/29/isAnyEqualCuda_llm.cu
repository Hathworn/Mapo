#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void isAnyEqualCuda(const uint8_t *image, uint8_t *value, size_t valueCount, uint32_t width, uint32_t height, uint32_t *differenceCount)
{
    // Calculate the unique identifier for each thread
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if the thread is within the image boundaries
    if (x < width && y < height) {
        const uint32_t id = y * width + x;

        // Use shared memory for value array to reduce memory access latency
        __shared__ uint8_t sharedValue[256];
        if (threadIdx.x < valueCount) {
            sharedValue[threadIdx.x] = value[threadIdx.x];
        }
        __syncthreads();

        bool equal = false;

        // Optimize loop with shared memory
        for (uint32_t i = 0; i < valueCount; ++i) {
            if (image[id] == sharedValue[i]) {
                equal = true;
                break;
            }
        }

        // Use atomic add to safely increment the count of differences
        if (equal)
            atomicAdd(differenceCount, 1);
    }
}