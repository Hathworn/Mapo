#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void isEqualCuda(const uint8_t* in1, uint32_t rowSizeIn1, const uint8_t* in2, uint32_t rowSizeIn2, uint32_t width, uint32_t height, uint32_t* isEqual) 
{
    // Use 1D block and grid for efficient memory access
    const uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t totalThreads = blockDim.x * gridDim.x;

    uint32_t localIsEqual = 1; // Local flag for partial results

    for (uint32_t i = idx; i < width * height; i += totalThreads) {
        const uint32_t y = i / width; // Calculate row
        const uint32_t x = i % width; // Calculate column
        if (in1[y * rowSizeIn1 + x] != in2[y * rowSizeIn2 + x]) {
            localIsEqual = 0; // If any element is not equal, set local flag to 0
        }
    }

    // Atomic AND on global flag only if a mismatch is found
    if(localIsEqual == 0)
        atomicAnd(isEqual, localIsEqual);
}