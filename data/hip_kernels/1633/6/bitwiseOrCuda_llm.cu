#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseOrCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height) {
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Check if within bounds
    if (x < width && y < height) {
        // Calculate linear offsets once per thread
        const uint32_t baseIndexIn1 = y * rowSizeIn1;
        const uint32_t baseIndexIn2 = y * rowSizeIn2;
        const uint32_t baseIndexOut = y * rowSizeOut;
        
        out[baseIndexOut + x] = in1[baseIndexIn1 + x] | in2[baseIndexIn2 + x];
    }
}