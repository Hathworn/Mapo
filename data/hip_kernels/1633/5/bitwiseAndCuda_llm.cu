#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseAndCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Cache block and thread index values for reuse
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Only compute if within bounds
    if (x < width && y < height) {
        // Compute linear index once
        const uint32_t rowIndex1 = y * rowSizeIn1;
        const uint32_t rowIndex2 = y * rowSizeIn2;
        const uint32_t rowIndexOut = y * rowSizeOut;
        
        // Perform bitwise and operation
        out[rowIndexOut + x] = in1[rowIndex1 + x] & in2[rowIndex2 + x];
    }
}