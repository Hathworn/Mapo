#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void invertCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global thread ID
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread coordinates are within image bounds
    if (x < width && y < height) {
        // Compute linear index for row-major order access and invert pixel
        const uint32_t indexIn = y * rowSizeIn + x;
        const uint32_t indexOut = y * rowSizeOut + x;
        out[indexOut] = ~in[indexIn];
    }
}