#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lookupTableCuda(const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height, uint8_t * table)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure the position is within bounds before accessing data
    if (x < width && y < height) {
        uint32_t in_index = y * rowSizeIn + x;    // Calculate input index
        uint32_t out_index = y * rowSizeOut + x;  // Calculate output index
        out[out_index] = table[in[in_index]];     // Apply lookup table
    }
}