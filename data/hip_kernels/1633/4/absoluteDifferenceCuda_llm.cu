#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void absoluteDifferenceCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        // Directly calculate offset without extra pointer arithmetic
        const uint32_t offsetIn1 = y * rowSizeIn1 + x;
        const uint32_t offsetIn2 = y * rowSizeIn2 + x;
        const uint32_t offsetOut = y * rowSizeOut + x;

        // Use register for temporary storage
        const uint8_t val1 = in1[offsetIn1];
        const uint8_t val2 = in2[offsetIn2];

        // Write result directly to output
        out[offsetOut] = (val1 > val2) ? (val1 - val2) : (val2 - val1);
    }
}