#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rotateCuda(const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, float inXStart, float inYStart, uint32_t width, uint32_t height, float cosAngle, float sinAngle)
{
    uint32_t outX = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t outY = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if the thread is within the output bounds
    if (outX < width && outY < height) {
        // Calculate exact input coordinates
        float exactInX = inXStart + cosAngle * outX + sinAngle * outY;
        float exactInY = inYStart - sinAngle * outX + cosAngle * outY;
        
        int32_t inX = static_cast<int32_t>(exactInX);
        int32_t inY = static_cast<int32_t>(exactInY);

        // Optimize thread output calculation
        uint32_t outIndex = outY * rowSizeOut + outX;

        // Ensure input coordinates are within bounds
        if (inX < 0 || inX >= width - 1 || inY < 0 || inY >= height - 1) {
            out[outIndex] = 0;
        } else {
            // Calculate input index and optimize bilinear interpolation
            uint32_t inIndex = inY * rowSizeIn + inX;
            float probX = exactInX - inX;
            float probY = exactInY - inY;

            float mean = in[inIndex] * (1 - probX) * (1 - probY) +
                         in[inIndex + 1] * probX * (1 - probY) +
                         in[inIndex + rowSizeIn] * (1 - probX) * probY +
                         in[inIndex + rowSizeIn + 1] * probX * probY + 0.5f;

            out[outIndex] = static_cast<uint8_t>(mean);
        }
    }
}