#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flipCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height, bool horizontal, bool vertical)
{
    const uint32_t inX = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t inY = blockDim.y * blockIdx.y + threadIdx.y;

    // Use shared memory for better memory access coalescing
    extern __shared__ uint8_t sharedMem[];

    if (inX < width && inY < height) {
        const uint32_t outX = horizontal ? (width - 1 - inX) : inX;
        const uint32_t outY = vertical ? (height - 1 - inY) : inY;

        // Load input data to shared memory
        sharedMem[threadIdx.y * blockDim.x + threadIdx.x] = in[inY * rowSizeIn + inX];
        __syncthreads();

        // Write output from shared memory
        out[outY * rowSizeOut + outX] = sharedMem[threadIdx.y * blockDim.x + threadIdx.x];
    }
}