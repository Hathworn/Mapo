#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void projectionProfileHorizontalCuda( const uint8_t * image, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * projection )
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure block-level atomic operations using shared memory
    __shared__ uint32_t shared_projection[1024]; // Assuming max blockDim.x of 1024
    if (threadIdx.y == 0)
        shared_projection[threadIdx.x] = 0;
    __syncthreads();

    if (x < width && y < height) {
        const uint8_t * imageX = image + y * rowSize + x;
        atomicAdd(&shared_projection[threadIdx.x], (*imageX));
    }
    __syncthreads();

    // Global atomic add only by the first thread in each block
    if (threadIdx.y == 0)
        atomicAdd(&projection[x], shared_projection[threadIdx.x]);
}