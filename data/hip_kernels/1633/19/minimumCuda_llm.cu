#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void minimumCuda(const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate single linear index to optimize memory coalescing
    const uint32_t idx = blockIdx.y * blockDim.y * rowSizeOut + blockIdx.x * blockDim.x + threadIdx.y * rowSizeOut + threadIdx.x;

    // Calculate row and column index
    const uint32_t x = idx % rowSizeOut;
    const uint32_t y = idx / rowSizeOut;

    if (x < width && y < height) {
        const uint8_t *in1X = in1 + y * rowSizeIn1 + x;
        const uint8_t *in2X = in2 + y * rowSizeIn2 + x;
        uint8_t *outX = out + y * rowSizeOut + x;
        (*outX) = ((*in1X) < (*in2X)) ? (*in1X) : (*in2X);
    }
}