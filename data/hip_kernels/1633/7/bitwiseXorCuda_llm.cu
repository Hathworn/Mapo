#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bitwiseXorCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
    // Calculate global thread indices
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds and perform XOR operation only if within valid range
    if ( x < width && y < height ) {
        const uint32_t idIn1 = y * rowSizeIn1 + x;
        const uint32_t idIn2 = y * rowSizeIn2 + x;
        const uint32_t idOut = y * rowSizeOut + x;
        
        // Perform bitwise XOR
        out[idOut] = in1[idIn1] ^ in2[idIn2];
    }
}