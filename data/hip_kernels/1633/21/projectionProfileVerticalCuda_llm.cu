#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void projectionProfileVerticalCuda(const uint8_t *image, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t *projection)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    // Use y as a thread-local accumulator to minimize atomic operations
    const uint32_t yStart = blockDim.y * blockIdx.y + threadIdx.y;
    const uint32_t stride = gridDim.y * blockDim.y; // Move down by entire grid height

    if (x < width) {
        uint32_t sum = 0;
        for (uint32_t y = yStart; y < height; y += stride) {
            const uint8_t *imageY = image + y * rowSize + x;
            sum += *imageY;
        }
        // Sum computed, perform atomic add once per thread for this x coordinate
        atomicAdd(&projection[yStart], sum);
    }
}