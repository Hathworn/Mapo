#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtractCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate x and y using direct multiplication, avoiding repeated blockIdx and threadIdx computation
    const uint32_t idx = blockDim.x * blockDim.y * blockIdx.y + blockDim.x * blockIdx.x + threadIdx.y * blockDim.x + threadIdx.x;

    // Use flat index for bounds check to ensure the thread is within valid pixel range
    if (idx < width * height) {
        // Calculate y and x coordinates based on the flat index
        const uint32_t x = idx % width;
        const uint32_t y = idx / width;

        // Use more descriptive array access with direct addition, reducing pointer arithmetic complexity
        out[y * rowSizeOut + x] = (in1[y * rowSizeIn1 + x] > in2[y * rowSizeIn2 + x]) 
                                  ? (in1[y * rowSizeIn1 + x] - in2[y * rowSizeIn2 + x]) 
                                  : 0;
    }
}