#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load accuracy to shared memory for faster access
    __shared__ float shared_accuracy;
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy;
    }
    __syncthreads();

    // Perform division using shared memory
    if (threadIdx.x == 0) {
        *accuracy = shared_accuracy / N;
    }
}