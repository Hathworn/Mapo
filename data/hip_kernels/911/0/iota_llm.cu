#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iota(int const size, int *data, int const value)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;  // Added stride for processing larger data
    for (int i = idx; i < size; i += stride)  // Loop with stride to cover all elements
    {
        data[i] = i + value;  // Perform the update
    }
}