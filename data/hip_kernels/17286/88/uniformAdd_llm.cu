#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    __shared__ float uni;
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];

    unsigned int address = blockIdx.x * (blockDim.x * 2) + baseIndex + threadIdx.x; // Simplified address calculation

    __syncthreads();

    // Optimize out-of-bounds check
    if (address < n)
        g_data[address] += uni;
    if (address + blockDim.x < n)
        g_data[address + blockDim.x] += uni;
}