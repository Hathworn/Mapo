#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Optimize by calculating global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we only access valid memory
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}