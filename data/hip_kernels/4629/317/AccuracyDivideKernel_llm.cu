#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load value into a register to reduce global memory accesses
    float acc = *accuracy;

    // Perform division
    acc /= N;

    // Write back the result to global memory only once
    *accuracy = acc;
}