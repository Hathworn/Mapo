#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define EMPTY_KEY_64 0  // Ensure EMPTY_KEY_64 is defined

__global__ void init_render_buffer(int64_t* render_buffer, const uint32_t qw_count) {
    const uint32_t index = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t stride = blockDim.x * gridDim.x;

    // Use index stride loop pattern for efficiency
    for (uint32_t i = index; i < qw_count; i += stride) {
        render_buffer[i] = EMPTY_KEY_64;
    }
}