#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void columnarize_groups(int8_t* columnar_buffer, const int8_t* rowwise_buffer, const size_t row_count, const size_t col_count, const size_t* col_widths, const size_t row_size) {
    // Calculate the thread index for current execution
    const auto thread_index = threadIdx.x + blockIdx.x * blockDim.x + blockIdx.y * blockDim.x * gridDim.x;

    // Return early if the thread index exceeds the row count
    if (thread_index >= row_count) {
        return;
    }

    auto read_ptr = rowwise_buffer + thread_index * row_size;
    auto col_base = columnar_buffer;

    // Use offsets to avoid multiple dereference operations
    size_t offset = 0;
    for (size_t i = 0; i < col_count; ++i) {
        const size_t col_width = col_widths[i];

        switch (col_width) {
            case 8: {
                // Use direct address calculation
                int64_t* write_ptr = reinterpret_cast<int64_t*>(col_base + offset) + thread_index;
                *write_ptr = *reinterpret_cast<const int64_t*>(read_ptr);
            } break;
            case 4: {
                // Use direct address calculation
                int32_t* write_ptr = reinterpret_cast<int32_t*>(col_base + offset) + thread_index;
                *write_ptr = *reinterpret_cast<const int32_t*>(read_ptr);
            } break;
            default:;
        }

        // Increment read and column base by the current column width
        offset += col_width * row_count;
        read_ptr += col_width;  // WARN(miyu): No padding!!
    }
}