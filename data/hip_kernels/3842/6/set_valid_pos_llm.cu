#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_valid_pos(int32_t* pos_buff, int32_t* count_buff, const int32_t entry_count) {
    const int32_t start = threadIdx.x + blockDim.x * blockIdx.x;
    const int32_t step = blockDim.x * gridDim.x;
    for (int32_t i = start; i < entry_count; i += step) {
        // Efficient memory access by avoiding redundant calculations
        int32_t pos_val = pos_buff[i];
        if (VALID_POS_FLAG == pos_val) {
            pos_buff[i] = (i == 0) ? 0 : count_buff[i - 1];
        }
    }
}