#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_group(int8_t* groups, const size_t group_count, const size_t col_count, const size_t* col_widths, const size_t* init_vals) {
    // Calculate global thread index
    const auto thread_index = blockIdx.x * blockDim.x + threadIdx.x;

    // Return if thread index is greater than or equal to group count
    if (thread_index >= group_count) {
        return;
    }

    // Optimize loop increments and data index calculations
    size_t offset = 0;
    for (size_t i = 0; i < col_count; ++i) {
        if (col_widths[i] == 4) {
            *reinterpret_cast<uint32_t*>(groups + offset + thread_index * col_widths[i]) = 
                *reinterpret_cast<const uint32_t*>(init_vals + i);
        } else if (col_widths[i] == 8) {
            reinterpret_cast<size_t*>(groups + offset)[thread_index] = init_vals[i];
        }
        offset += col_widths[i] * group_count; // Increase offset by col_width[i] * group_count
    }
}