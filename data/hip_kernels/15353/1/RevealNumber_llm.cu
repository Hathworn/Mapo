#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void RevealNumber(int* number, unsigned int number_size)
{
    // Optimize idx calculation to properly use block and grid
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < number_size)
    {
        // Efficiently print the index and number
        printf("Index: %u, Value: %i\n", idx, number[idx]);
    }
}