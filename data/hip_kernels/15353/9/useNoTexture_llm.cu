#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void useNoTexture(float* pin, float* pout, int len)
{
    // Calculate linear index directly for read and write operations
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idz = blockIdx.z * blockDim.z + threadIdx.z;
    unsigned int index = idx + c_size.x * (idy + idz * c_size.y);

    // Use shared memory for better memory access patterns
    extern __shared__ float shared_mem[];
    float* shared_a = shared_mem;
    float* shared_b = shared_mem + blockDim.x * blockDim.y * blockDim.z;
    float* shared_c = shared_mem + 2 * blockDim.x * blockDim.y * blockDim.z;

    shared_a[threadIdx.x + blockDim.x * (threadIdx.y + threadIdx.z * blockDim.y)] = pin[0 + len * index];
    shared_b[threadIdx.x + blockDim.x * (threadIdx.y + threadIdx.z * blockDim.y)] = pin[1 + len * index];
    shared_c[threadIdx.x + blockDim.x * (threadIdx.y + threadIdx.z * blockDim.y)] = pin[2 + len * index];
    __syncthreads();

    // Compute result using shared memory
    float a = shared_a[threadIdx.x + blockDim.x * (threadIdx.y + threadIdx.z * blockDim.y)];
    float b = shared_b[threadIdx.x + blockDim.x * (threadIdx.y + threadIdx.z * blockDim.y)];
    float c = shared_c[threadIdx.x + blockDim.x * (threadIdx.y + threadIdx.z * blockDim.y)];

    // Store the result back into global memory
    pout[index] = sqrtf(powf(a, 2) + powf(b, 2) + powf(c, 2));
}