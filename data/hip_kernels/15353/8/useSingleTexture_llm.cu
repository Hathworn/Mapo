#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void useSingleTexture(hipTextureObject_t tex, float* pout)
{
    // Precompute indices
    unsigned int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int globalIdy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int globalIdz = blockIdx.z * blockDim.z + threadIdx.z;
    unsigned int linearIdx = globalIdx + c_size.x * (globalIdy + globalIdz * c_size.y);

    // Sample the texture
    float4 sample = tex3D<float4>(tex, globalIdx + 0.5f, globalIdy + 0.5f, globalIdz + 0.5f);

    // Store result in output array using square root of dot product
    pout[linearIdx] = sqrtf(sample.x * sample.x + sample.y * sample.y + sample.z * sample.z);
}