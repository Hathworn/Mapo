#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float* indata, float* outdata, int* Size, float* Spacing)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Return early if out of bounds
    if (i >= Size[0] || j >= Size[1]) {
        return;
    }

    // Calculate linear index variable once
    int index = j + i * Size[1];
    
    // Compute output data with pre-computed index
    outdata[index] = indata[index] * (Size[0] + Spacing[1]);

    // Minimize print overhead by avoiding repeated calculations in printf
    printf("[%d,%d] -> %.2f -> %.2f\n", i, j, indata[index], outdata[index]);
}