#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeKernel(float* vec, int len)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Early exit for out-of-bound threads for efficiency
    if (i >= c_size.x || j >= c_size.y || k >= c_size.z)
        return;

    // Pre-calculate indices to reduce redundant computation
    long int base_id = len * (i + c_size.x * (j + k * c_size.y));
    
    // Use temporary variable for loop index to prevent redundant recalculation
    for (long int w = 0; w < len; ++w)
    {
        long int id = w + base_id;
        vec[id] = id;
    }
}