#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_scatter_cu_z(double *c, const double *pwcc, const double scale, const int ngpts, const int nmaps, const int *ghatmap) {

    // Calculate global thread index for enhanced readability
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x * blockIdx.y * gridDim.x;

    // Ensure thread is within bounds
    if (igpt < ngpts) {
        // Vectorized access for better memory coalescing
        int index1 = 2 * igpt;
        int index2 = 2 * ghatmap[igpt];

        // Write scaled values into output arrays
        c[index2] = scale * pwcc[index1];
        c[index2 + 1] = scale * pwcc[index1 + 1];

        // Additional mapping if nmaps is 2
        if (nmaps == 2) {
            int index3 = 2 * ghatmap[igpt + ngpts];
            c[index3] = scale * pwcc[index1];
            c[index3 + 1] = -scale * pwcc[index1 + 1];
        }
    }
}