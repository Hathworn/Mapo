#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_copy_rc_cu_z(const double *din, double *zout, const int n) {
    // Calculate global thread ID using 1D block and grid
    int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    if (igpt < n) {
        // Copy input to output and initialize the imaginary part to zero
        zout[2 * igpt] = din[igpt];
        zout[2 * igpt + 1] = 0.0;
    }
}