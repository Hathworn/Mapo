#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_copy_cr_cu_z(const double *zin, double *dout, const int n) {
    int igpt = blockIdx.x * blockDim.x + threadIdx.x; // Compute the thread's unique index

    for (int idx = igpt; idx < n; idx += blockDim.x * gridDim.x) {
        dout[idx] = zin[2 * idx]; // Copy and stride over zin
    }
}