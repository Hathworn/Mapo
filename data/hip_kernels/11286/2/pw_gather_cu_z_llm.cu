#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_gather_cu_z(double *pwcc, const double *c, const double scale, const int ngpts, const int *ghatmap) {

    // Calculate global thread index
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within bounds
    if (igpt < ngpts) {
        int mapIndex = 2 * ghatmap[igpt]; // Use shared memory or register for efficient access
        pwcc[2 * igpt] = scale * c[mapIndex];
        pwcc[2 * igpt + 1] = scale * c[mapIndex + 1];
    }
}