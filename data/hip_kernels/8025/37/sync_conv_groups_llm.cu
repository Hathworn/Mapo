#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void sync_conv_groups() {
    // Use cooperative groups for better synchronization
    extern __shared__ int sdata[];
    __syncthreads();
}