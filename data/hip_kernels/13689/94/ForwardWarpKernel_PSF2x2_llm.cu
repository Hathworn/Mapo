#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    // Calculate the global x and y index for the thread
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Return early if out of bounds
    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    int tx = (int) px;
    int ty = (int) py;

    float value = src[image_row_offset + j];
    float weight;

    // Precompute weights
    float w00 = (1.0f - dx) * (1.0f - dy);
    float w01 = dx * (1.0f - dy);
    float w10 = (1.0f - dx) * dy;
    float w11 = dx * dy;

    // Fill pixel containing bottom right corner
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        weight = w11;
        atomicAdd(&dst[ty * image_stride + tx], value * weight);
        atomicAdd(&normalization_factor[ty * image_stride + tx], weight);
    }

    // Fill pixel containing bottom left corner
    if (tx - 1 < w && tx - 1 >= 0 && ty < h && ty >= 0)
    {
        weight = w10;
        atomicAdd(&dst[ty * image_stride + (tx - 1)], value * weight);
        atomicAdd(&normalization_factor[ty * image_stride + (tx - 1)], weight);
    }

    // Fill pixel containing upper left corner
    if (tx - 1 < w && tx - 1 >= 0 && ty - 1 < h && ty - 1 >= 0)
    {
        weight = w00;
        atomicAdd(&dst[(ty - 1) * image_stride + (tx - 1)], value * weight);
        atomicAdd(&normalization_factor[(ty - 1) * image_stride + (tx - 1)], weight);
    }

    // Fill pixel containing upper right corner
    if (tx < w && tx >= 0 && ty - 1 < h && ty - 1 >= 0)
    {
        weight = w01;
        atomicAdd(&dst[(ty - 1) * image_stride + tx], value * weight);
        atomicAdd(&normalization_factor[(ty - 1) * image_stride + tx], weight);
    }
}