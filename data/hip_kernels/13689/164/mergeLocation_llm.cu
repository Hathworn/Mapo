#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread execution only if within bounds
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];

        // Load, compute and store efficiently
        float scaled_x = __fmul_rn(loc.x, scale);
        float scaled_y = __fmul_rn(loc.y, scale);
        
        x[ptidx] = scaled_x;
        y[ptidx] = scaled_y;
    }
}