#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global thread index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Use one-dimensional thread indexing to reduce calculations
    int pos = i * w + j;

    // Ensure thread index is within bounds
    if (pos < w * h)
    {
        image[pos] = value; // Assign value to image
    }
}