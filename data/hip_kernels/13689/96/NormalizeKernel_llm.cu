#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilize shared memory to improve performance
    __shared__ float shared_normalization_factor[32][32]; // Adjust size based on maximum block size

    if (i < h && j < w) {
        const int pos = i * s + j;

        shared_normalization_factor[threadIdx.y][threadIdx.x] = normalization_factor[pos]; // Load to shared memory

        __syncthreads(); // Synchronize to ensure all threads have loaded their data

        float scale = shared_normalization_factor[threadIdx.y][threadIdx.x]; // Access from shared memory

        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

        image[pos] *= invScale;
    }
}