#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Create a temporary variable to prevent global memory write conflicts
    float temp_accuracy = *accuracy; 
    temp_accuracy /= N;
    *accuracy = temp_accuracy; // Update the original value after computation
}