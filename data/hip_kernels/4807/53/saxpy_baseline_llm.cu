#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_baseline(float* y, float* x, float a, clock_t * timer_vals)
{
    // Calculate the globally unique thread index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Efficiently handle NUM_ITERS computation for each thread
    for (int i = 0; i < NUM_ITERS; i++) {
        unsigned int offset_idx = i * blockDim.x * gridDim.x + idx;
        y[offset_idx] = a * x[offset_idx] + y[offset_idx];
    }
}