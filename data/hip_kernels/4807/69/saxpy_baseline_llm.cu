#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_optimized(float* y, float* x, float a, clock_t* timer_vals) {
    // Calculate global index once
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x; 
    unsigned int stride = blockDim.x * gridDim.x;
    
    // Loop with stride to handle more elements
    for (int i = 0; i < NUM_ITERS; i++) {
        unsigned int idx = i * stride + tid;
        y[idx] = a * x[idx] + y[idx];
    }
}