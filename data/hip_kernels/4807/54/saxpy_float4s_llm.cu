#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_float4s(float* y, float* x, float a, clock_t* timer_vals)
{
    int idx = (blockIdx.x * blockDim.x + threadIdx.x); // Compute global index based on block and thread index
    int stride = gridDim.x * blockDim.x; // Calculate total number of threads across the grid

    float4* x_as_float4 = (float4*)x;
    float4* y_as_float4 = (float4*)y;

    for (int i = idx; i < NUM_ITERS / 4; i += stride) {
        float4 tmp1_x = x_as_float4[i];
        float4 tmp1_y = y_as_float4[i];

        // Combine computation into a single float4 operation
        y_as_float4[i] = make_float4(
            a * tmp1_x.x + tmp1_y.x,
            a * tmp1_x.y + tmp1_y.y,
            a * tmp1_x.z + tmp1_y.z,
            a * tmp1_x.w + tmp1_y.w);
    }
}