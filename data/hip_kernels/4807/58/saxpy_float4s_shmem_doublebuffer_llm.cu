#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_float4s_shmem_doublebuffer(float* y, float* x, float a, clock_t* timer_vals) {
    // Use float4 for shared memory to improve coalescence
    extern __shared__ float4 shared_mem[];
    float4* sdata_x0 = shared_mem;
    float4* sdata_x1 = shared_mem + COMPUTE_THREADS_PER_CTA;

    int tid = threadIdx.x;
    unsigned int idx0 = blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;
    unsigned int idx1 = COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;

    float4* x_as_float4 = (float4*)x;
    float4* y_as_float4 = (float4*)y;
    float4 result_y;

    for (int i = 0; i < NUM_ITERS / 4; i += 2) {
        __syncthreads();
        float4 tmp1_x = x_as_float4[idx0];
        float4 tmp1_y = y_as_float4[idx0];

        if (i != 0) {
            result_y.x = a * sdata_x1[tid].x + sdata_x1[tid].y;
            result_y.y = a * sdata_x1[tid].z + sdata_x1[tid].w;
            result_y.z = a * sdata_x1[tid].x + sdata_x1[tid].y;
            result_y.w = a * sdata_x1[tid].z + sdata_x1[tid].w;
            y_as_float4[idx1] = result_y;
            idx1 += 2 * COMPUTE_THREADS_PER_CTA * CTA_COUNT;
        }

        // Store float4 to shared memory directly
        sdata_x0[tid] = tmp1_x;
        __syncthreads();

        tmp1_x = x_as_float4[idx1];
        tmp1_y = y_as_float4[idx1];
        
        result_y.x = a * sdata_x0[tid].x + sdata_x0[tid].y;
        result_y.y = a * sdata_x0[tid].z + sdata_x0[tid].w;
        result_y.z = a * sdata_x0[tid].x + sdata_x0[tid].y;
        result_y.w = a * sdata_x0[tid].z + sdata_x0[tid].w;
        
        y_as_float4[idx0] = result_y;
        idx0 += 2 * COMPUTE_THREADS_PER_CTA * CTA_COUNT;
        
        sdata_x1[tid] = tmp1_x; // Use second buffer for next iteration
    }
    
    __syncthreads();
    // Final writeback in case loop ends before updating idx1
    result_y.x = a * sdata_x1[tid].x + sdata_x1[tid].y;
    result_y.y = a * sdata_x1[tid].z + sdata_x1[tid].w;
    result_y.z = a * sdata_x1[tid].x + sdata_x1[tid].y;
    result_y.w = a * sdata_x1[tid].z + sdata_x1[tid].w;
    
    y_as_float4[idx1] = result_y;
}