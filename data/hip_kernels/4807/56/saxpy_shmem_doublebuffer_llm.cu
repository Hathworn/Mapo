#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_shmem_doublebuffer (float* y, float* x, float a, clock_t *timer_vals)
{
    extern __shared__ float sdata[];
    float* sdata_x0 = sdata;
    float* sdata_y0 = sdata + COMPUTE_THREADS_PER_CTA;
    float* sdata_x1 = sdata + 2 * COMPUTE_THREADS_PER_CTA;
    float* sdata_y1 = sdata + 3 * COMPUTE_THREADS_PER_CTA;

    int tid = threadIdx.x;
    unsigned int idx0 = blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;
    unsigned int idx1 = COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;

    for (int i = 0; i < NUM_ITERS; i += 2) {
        __syncthreads();

        // Load data into shared memory in coalesced manner
        if (tid < COMPUTE_THREADS_PER_CTA) {
            sdata_x0[tid] = x[idx0];
            sdata_y0[tid] = y[idx0];
        }
        __syncthreads();

        if (i != 0) {
            // Perform SAXPY operation and store result
            y[idx1] = a * sdata_x1[tid] + sdata_y1[tid];
            idx1 += 2 * COMPUTE_THREADS_PER_CTA * CTA_COUNT;
        }
        __syncthreads();

        // Load next set of data into shared memory in coalesced manner
        if (tid < COMPUTE_THREADS_PER_CTA) {
            sdata_x1[tid] = x[idx1];
            sdata_y1[tid] = y[idx1];
        }
        __syncthreads();

        // Perform SAXPY operation and store result
        y[idx0] = a * sdata_x0[tid] + sdata_y0[tid];
        idx0 += 2 * COMPUTE_THREADS_PER_CTA * CTA_COUNT;
    }

    __syncthreads();
    // Final write-back for last iteration
    y[idx1] = a * sdata_x1[tid] + sdata_y1[tid];
}