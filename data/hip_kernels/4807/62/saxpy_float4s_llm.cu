#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_float4s (float* y, float* x, float a, clock_t *timer_vals)
{
    // Using float4 pointers to handle four elements per thread improve memory operations.
    float4 * x_as_float4 = reinterpret_cast<float4 *>(x);
    float4 * y_as_float4 = reinterpret_cast<float4 *>(y);

    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    // Loop unrolling for better performance
    for (int i = idx; i < NUM_ITERS/4; i += stride) {
        float4 tmp1_x = x_as_float4[i];
        float4 tmp1_y = y_as_float4[i];

        // Vectorized operation for performance
        float4 result_y;
        result_y.x = a * tmp1_x.x + tmp1_y.x;
        result_y.y = a * tmp1_x.y + tmp1_y.y;
        result_y.z = a * tmp1_x.z + tmp1_y.z;
        result_y.w = a * tmp1_x.w + tmp1_y.w;
        
        y_as_float4[i] = result_y;
    }
}