#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemvn_kernel2_fermi(int n, int m, int n1, float alpha,  float* A, int lda, float *x, float *y)
{
    int ind = blockIdx.x * num_threads + threadIdx.x;

    A += ind;
    x += threadIdx.x;

    float res = 0.f;

    __shared__ float buff[num_threads];
    for(int i = 0; i < n1; i += num_threads) {
        // Load shared memory with x values
        if (threadIdx.x < num_threads && (i + threadIdx.x) < m) {
            buff[threadIdx.x] = x[i + threadIdx.x];
        }
        __syncthreads();

        #pragma unroll
        for(int j = 0; j < num_threads; j++) {
            res += A[0] * buff[j];
            A += lda;
        }
        __syncthreads();
    }
    
    // Final result for output y
    if (ind < n) {
        y[ind] = alpha * res;
    }
}