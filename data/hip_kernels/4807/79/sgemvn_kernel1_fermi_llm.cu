#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sgemvn_kernel1_fermi(int n, int m, int n1, float alpha, float* A, int lda, float *x, float *y)
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x for better readability

    if (ind >= n) return; // Early exit if index is out of bounds

    A += ind;
    float res = 0.f;
    
    int i = 0;
    for (; i < n1; i += sgemv_bs) {
        #pragma unroll
        for (int j = 0; j < sgemv_bs; j++) {
            res += A[0] * x[j];
            A += lda;
        }
        x += sgemv_bs;
    }

    // Remove the disabled code section to reduce clutter

    y[ind] = alpha * res; // Direct assignment
}