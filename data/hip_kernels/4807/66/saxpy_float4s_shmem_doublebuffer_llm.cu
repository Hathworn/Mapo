#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_float4s_shmem_doublebuffer(float* y, float* x, float a, clock_t *timer_vals)
{
    // Use a single shared memory buffer with 2 float4 arrays for double buffering
    extern __shared__ float4 sdata[];
    float4 *sdata_x_0 = sdata;
    float4 *sdata_y_0 = sdata + COMPUTE_THREADS_PER_CTA;
    float4 *sdata_x_1 = sdata + 2 * COMPUTE_THREADS_PER_CTA;
    float4 *sdata_y_1 = sdata + 3 * COMPUTE_THREADS_PER_CTA;
    
    int tid = threadIdx.x;
    unsigned int idx0 = blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;
    unsigned int idx1 = COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;
    
    float4 *x_as_float4 = (float4 *)x;
    float4 *y_as_float4 = (float4 *)y;
    float4 result_y;
    
    for (int i = 0; i < NUM_ITERS / 4; i += 2) {
        // Load x and y to shared memory
        float4 tmp1_x = x_as_float4[idx0];
        float4 tmp1_y = y_as_float4[idx0];
        
        __syncthreads();
        
        if (i != 0) {
            // Compute and store result from previous iteration
            result_y.x = a * sdata_x_1[tid].x + sdata_y_1[tid].x;
            result_y.y = a * sdata_x_1[tid].y + sdata_y_1[tid].y;
            result_y.z = a * sdata_x_1[tid].z + sdata_y_1[tid].z;
            result_y.w = a * sdata_x_1[tid].w + sdata_y_1[tid].w;
            y_as_float4[idx1] = result_y;
            idx1 += 2 * COMPUTE_THREADS_PER_CTA * CTA_COUNT;
        }
        
        sdata_x_0[tid] = tmp1_x;
        sdata_y_0[tid] = tmp1_y;
        
        __syncthreads();
        
        // Process next set of data
        tmp1_x = x_as_float4[idx1];
        tmp1_y = y_as_float4[idx1];
        result_y.x = a * sdata_x_0[tid].x + sdata_y_0[tid].x;
        result_y.y = a * sdata_x_0[tid].y + sdata_y_0[tid].y;
        result_y.z = a * sdata_x_0[tid].z + sdata_y_0[tid].z;
        result_y.w = a * sdata_x_0[tid].w + sdata_y_0[tid].w;
        y_as_float4[idx0] = result_y;
        idx0 += 2 * COMPUTE_THREADS_PER_CTA * CTA_COUNT;
        
        sdata_x_1[tid] = tmp1_x;
        sdata_y_1[tid] = tmp1_y;
    }
    
    __syncthreads();
    // Final computation for leftover data
    result_y.x = a * sdata_x_1[tid].x + sdata_y_1[tid].x;
    result_y.y = a * sdata_x_1[tid].y + sdata_y_1[tid].y;
    result_y.z = a * sdata_x_1[tid].z + sdata_y_1[tid].z;
    result_y.w = a * sdata_x_1[tid].w + sdata_y_1[tid].w;
    y_as_float4[idx1] = result_y;
}