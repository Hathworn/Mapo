#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void saxpy_float4s_shmem(float* y, float* x, float a, clock_t* timer_vals) 
{
    // Use float4 for shared memory to load/store data more efficiently.
    __shared__ float4 sdata_x[COMPUTE_THREADS_PER_CTA];
    __shared__ float4 sdata_y[COMPUTE_THREADS_PER_CTA];
    
    int tid = threadIdx.x;

    for (int i = 0; i < NUM_ITERS / 4; i++) {
        unsigned int idx = i * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;
        
        // Synchronize threads to ensure all data is loaded before proceeding.
        __syncthreads();
        
        // Cast global memory data to float4 and load into shared memory.
        float4* x_as_float4 = (float4*)x;
        float4* y_as_float4 = (float4*)y;
        
        float4 tmp_x = x_as_float4[idx];
        float4 tmp_y = y_as_float4[idx];
        
        // Store into shared memory.
        sdata_x[tid] = tmp_x;
        sdata_y[tid] = tmp_y;
        
        // Synchronize again before doing the computation.
        __syncthreads();
        
        // Perform SAXPY operation with shared memory data.
        float4 result_y;
        result_y.x = a * sdata_x[tid].x + sdata_y[tid].x;
        result_y.y = a * sdata_x[tid].y + sdata_y[tid].y;
        result_y.z = a * sdata_x[tid].z + sdata_y[tid].z;
        result_y.w = a * sdata_x[tid].w + sdata_y[tid].w;
        
        // Store result back to global memory.
        y_as_float4[idx] = result_y;
    }
}