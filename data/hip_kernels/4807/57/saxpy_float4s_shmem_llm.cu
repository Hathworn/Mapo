#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy_float4s_shmem (float* y, float* x, float a, clock_t * timer_vals)
{
    // Use float4 shared memory to utilize vectorized operations 
    extern __shared__ float4 sdata[];

    int tid = threadIdx.x;

    for (int i = 0; i < NUM_ITERS / 4; i++) {
        unsigned int idx = i * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + threadIdx.x;

        // Sync threads before loading data to shared memory
        __syncthreads();

        float4* x_as_float4 = (float4*)x;
        float4* y_as_float4 = (float4*)y;
        
        // Load x and y into shared memory using float4
        sdata[tid] = x_as_float4[idx];
        float4 tmp1_y = y_as_float4[idx];

        // Sync threads to ensure all data is loaded
        __syncthreads();

        // Perform computation using shared memory
        float4 result_y;
        result_y.x = a * sdata[tid].x + tmp1_y.x;
        result_y.y = a * sdata[tid].y + tmp1_y.y;
        result_y.z = a * sdata[tid].z + tmp1_y.z;
        result_y.w = a * sdata[tid].w + tmp1_y.w;

        // Store result
        y_as_float4[idx] = result_y;
    }
}