#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void saxpy_shmem (float* y, float* x, float a, clock_t * timer_vals)
{
    // Use shared memory to optimize data loading and computation
    __shared__ float sdata_x0[COMPUTE_THREADS_PER_CTA];
    __shared__ float sdata_y0[COMPUTE_THREADS_PER_CTA];
    int tid = threadIdx.x;
    
    for (int i = 0; i < NUM_ITERS; ++i) {
        unsigned int idx = i * COMPUTE_THREADS_PER_CTA * CTA_COUNT + blockIdx.x * COMPUTE_THREADS_PER_CTA + tid;
        
        // Load data into shared memory more efficiently
        __syncthreads();
        sdata_x0[tid] = x[idx];
        sdata_y0[tid] = y[idx];
        __syncthreads();
        
        // Compute SAXPY in shared memory
        y[idx] = a * sdata_x0[tid] + sdata_y0[tid];
    }
}