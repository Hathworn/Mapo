#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ size_t gpu_scalar_index(unsigned int x, unsigned int y, unsigned int z)
{
    return NX * (NY * z + y) + x;
}

__global__ void gpu_efield(double *fi, double *ex, double *ey, double *ez) {
    unsigned int y = blockIdx.y;
    unsigned int z = blockIdx.z;
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Precompute commonly used indices to reduce computation overhead
    size_t idx = gpu_scalar_index(x, y, z);
    
    unsigned int xp1 = (x + 1) % NX;
    unsigned int yp1 = (y + 1) % NY;
    unsigned int zp1 = (z + 1) % NZ;

    unsigned int xm1 = (NX + x - 1) % NX;
    unsigned int ym1 = (NY + y - 1) % NY;
    unsigned int zm1 = (NZ + z - 1) % NZ;
    
    // Reduce repeated function calls by storing results in temporary variables
    double fi_xm1 = fi[gpu_scalar_index(xm1, y, z)];
    double fi_xp1 = fi[gpu_scalar_index(xp1, y, z)];
    double fi_ym1 = fi[gpu_scalar_index(x, ym1, z)];
    double fi_yp1 = fi[gpu_scalar_index(x, yp1, z)];
    double fi_zm1 = fi[gpu_scalar_index(x, y, zm1)];
    double fi_zp1 = fi[gpu_scalar_index(x, y, zp1)];

    ex[idx] = 0.5 * (fi_xm1 - fi_xp1) / dx;
    ey[idx] = 0.5 * (fi_ym1 - fi_yp1) / dy;
    ez[idx] = 0.5 * (fi_zm1 - fi_zp1) / dz;
}