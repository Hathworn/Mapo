#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ size_t gpu_fieldn_index(unsigned int x, unsigned int y, unsigned int z, unsigned int d)
{
    return (NX*(NY*(NZ*(d-1)+z)+y)+x);
}

__global__ void gpu_stream(double *f0, double *f1, double *f2, double *h0, double *h1, double *h2, double *temp0, double *temp1, double *temp2)
{
    unsigned int y = blockIdx.y;
    unsigned int z = blockIdx.z;
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;

    // Streaming step
    unsigned int xp1 = (x + 1) % NX;
    unsigned int yp1 = (y + 1) % NY;
    unsigned int zp1 = (z + 1) % NZ;
    unsigned int xm1 = (NX + x - 1) % NX;
    unsigned int ym1 = (NY + y - 1) % NY;
    unsigned int zm1 = (NZ + z - 1) % NZ;

    #pragma unroll 27
    for (unsigned int d = 1; d <= 26; ++d) {
        // Precompute source indices based on direction `d`
        size_t src_index_f = 0, src_index_h = 0, src_index_temp = 0;

        // Use a single switch to precompute these indices
        switch(d) {
            case 1: src_index_f = gpu_fieldn_index(xm1, y, z, 1); break;
            case 2: src_index_f = gpu_fieldn_index(xp1, y, z, 2); break;
            case 3: src_index_f = gpu_fieldn_index(x, ym1, z, 3); break;
            case 4: src_index_f = gpu_fieldn_index(x, yp1, z, 4); break;
            case 5: src_index_f = gpu_fieldn_index(x, y, zm1, 5); break;
            case 6: src_index_f = gpu_fieldn_index(x, y, zp1, 6); break;
            case 7: src_index_f = gpu_fieldn_index(xm1, ym1, z, 7); break;
            case 8: src_index_f = gpu_fieldn_index(xp1, yp1, z, 8); break;
            case 9: src_index_f = gpu_fieldn_index(xm1, y, zm1, 9); break;
            case 10: src_index_f = gpu_fieldn_index(xp1, y, zp1, 10); break;
            case 11: src_index_f = gpu_fieldn_index(x, ym1, zm1, 11); break;
            case 12: src_index_f = gpu_fieldn_index(x, yp1, zp1, 12); break;
            case 13: src_index_f = gpu_fieldn_index(xm1, yp1, z, 13); break;
            case 14: src_index_f = gpu_fieldn_index(xp1, ym1, z, 14); break;
            case 15: src_index_f = gpu_fieldn_index(xm1, y, zp1, 15); break;
            case 16: src_index_f = gpu_fieldn_index(xp1, y, zm1, 16); break;
            case 17: src_index_f = gpu_fieldn_index(x, ym1, zp1, 17); break;
            case 18: src_index_f = gpu_fieldn_index(x, yp1, zm1, 18); break;
            case 19: src_index_f = gpu_fieldn_index(xm1, ym1, zm1, 19); break;
            case 20: src_index_f = gpu_fieldn_index(xp1, yp1, zp1, 20); break;
            case 21: src_index_f = gpu_fieldn_index(xm1, ym1, zp1, 21); break;
            case 22: src_index_f = gpu_fieldn_index(xp1, yp1, zm1, 22); break;
            case 23: src_index_f = gpu_fieldn_index(xm1, yp1, zm1, 23); break;
            case 24: src_index_f = gpu_fieldn_index(xp1, ym1, zp1, 24); break;
            case 25: src_index_f = gpu_fieldn_index(xp1, ym1, zm1, 25); break;
            case 26: src_index_f = gpu_fieldn_index(xm1, yp1, zp1, 26); break;
        }
        src_index_h = src_index_temp = src_index_f;  // shared computation for all arrays

        // Assign values for `f`, `h` and `temp`
        f1[gpu_fieldn_index(x, y, z, d)] = f2[src_index_f];
        h1[gpu_fieldn_index(x, y, z, d)] = h2[src_index_h];
        temp1[gpu_fieldn_index(x, y, z, d)] = temp2[src_index_temp];
    }
}