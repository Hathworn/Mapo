#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ size_t gpu_fieldn_index(unsigned int x, unsigned int y, unsigned int z, unsigned int d) {
    return (NX*(NY*(NZ*(d-1)+z)+y)+x);
}

__device__ __forceinline__ size_t gpu_scalar_index(unsigned int x, unsigned int y, unsigned int z) {
    return NX*(NY*z + y)+x;
}

__device__ __forceinline__ size_t gpu_field0_index(unsigned int x, unsigned int y, unsigned int z) {
    return NX*(NY*z + y)+x;
}

__global__ void gpu_init_equilibrium(double *f0, double *f1, double *h0, double *h1, double *temp0, double *temp1, double *r, double *c, double *u, double *v, double *w, double *ex, double *ey, double *ez, double *temp) {
    unsigned int y = blockIdx.y;
    unsigned int z = blockIdx.z;
    unsigned int x = blockIdx.x*blockDim.x+threadIdx.x;

    double rho = r[gpu_scalar_index(x,y,z)];
    double ux = u[gpu_scalar_index(x,y,z)];
    double uy = v[gpu_scalar_index(x,y,z)];
    double uz = w[gpu_scalar_index(x,y,z)];
    double charge = c[gpu_scalar_index(x,y,z)];
    double Ex = ex[gpu_scalar_index(x,y,z)];
    double Ey = ey[gpu_scalar_index(x,y,z)];
    double Ez = ez[gpu_scalar_index(x,y,z)];
    double Temp = temp[gpu_scalar_index(x,y,z)];

    // Temporary variables
    double w0r = w0 * rho;
    double wsr = ws * rho;
    double war = wa * rho;
    double wdr = wd * rho;

    double w0c = w0 * charge;
    double wsc = ws * charge;
    double wac = wa * charge;
    double wdc = wd * charge;

    double w0t = w0 * Temp;
    double wst = ws * Temp;
    double wat = wa * Temp;
    double wdt = wd * Temp;

    double omusq = 1.0 - 0.5 * (ux*ux+uy*uy+uz*uz)/cs_square;
    double omusq_c = 1.0 - 0.5 * ((ux + K*Ex)*(ux + K*Ex) + (uy + K*Ey)*(uy + K*Ey) + (uz + K*Ez)*(uz + K*Ez)) / cs_square;

    double tux = ux / cs_square / CFL;
    double tuy = uy / cs_square / CFL;
    double tuz = uz / cs_square / CFL;
    double tux_c = (ux + K*Ex) / cs_square / CFL;
    double tuy_c = (uy + K*Ey) / cs_square / CFL;
    double tuz_c = (uz + K*Ez) / cs_square / CFL;

    // Optimization by precomputing repeated expressions
    double omusq_factors[] = {tux, -tux, tuy, -tuy, tuz, -tuz, tux+tuy, -tuy-tux, tux+tuz, -tux-tuz, tuz+tuy, -tuy-tuz, tux-tuy, tuy-tux, tux-tuz, tuz-tux, tuy-tuz, tuz-tuy, tux+tuy+tuz, -tuy-tux-tuz, tux+tuy-tuz, tuz-(tux+tuy), tux+tuz-tuy, tuy-(tux+tuz), tuy+tuz-tux, tux-(tuy+tuz)};
    double factors_c[] = {tux_c, -tux_c, tuy_c, -tuy_c, tuz_c, -tuz_c, tux_c+tuy_c, -tuy_c-tux_c, tux_c+tuz_c, -tux_c-tuz_c, tuy_c+tuz_c, -tuy_c-tuz_c, tux_c-tuy_c, tuy_c-tux_c, tux_c-tuz_c, tuz_c-tux_c, tuy_c-tuz_c, tuz_c-tuy_c, tux_c+tuy_c+tuz_c, -tux_c-tuy_c-tuz_c, tux_c+tuy_c-tuz_c, tuz_c-(tux_c+tuy_c), tux_c+tuz_c-tuy_c, tuy_c-(tux_c+tuz_c), tuy_c+tuz_c-tux_c, tux_c-(tuy_c+tuz_c)};

    // Zero weight assignments
    f0[gpu_field0_index(x,y,z)] = w0r * omusq;
    h0[gpu_field0_index(x,y,z)] = w0c * omusq_c;
    temp0[gpu_field0_index(x, y, z)] = w0t * omusq;

    // Loop over directions to optimize repetitive code
    for (int i = 0; i < 6; ++i) {
        f1[gpu_fieldn_index(x,y,z,i+1)] = wsr * (omusq + omusq_factors[i]*(1.0 + 0.5*omusq_factors[i]));
        h1[gpu_fieldn_index(x,y,z,i+1)] = wsc * (omusq_c + factors_c[i]*(1.0 + 0.5*factors_c[i]));
        temp1[gpu_fieldn_index(x, y, z, i+1)] = wst * (omusq + omusq_factors[i]*(1.0 + 0.5*omusq_factors[i]));
    }
    
    for (int i = 6; i < 18; ++i) {
        f1[gpu_fieldn_index(x,y,z,i+1)] = ((i < 12) ? war : wdr) * (omusq + omusq_factors[i]*(1.0 + 0.5*omusq_factors[i]));
        h1[gpu_fieldn_index(x,y,z,i+1)] = ((i < 12) ? wac : wdc) * (omusq_c + factors_c[i]*(1.0 + 0.5*factors_c[i]));
        temp1[gpu_fieldn_index(x, y, z, i+1)] = ((i < 12) ? wat : wdt) * (omusq + omusq_factors[i]*(1.0 + 0.5*omusq_factors[i]));
    }
}