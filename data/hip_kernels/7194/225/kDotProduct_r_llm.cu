#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target,  const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];
    
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Unroll loop to improve performance
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        sum += a[eidx] * b[eidx];
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();
    
    // Use loop for warp reduction instead of multiple if-statements
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            shmem[threadIdx.x] += shmem[threadIdx.x + s];
        }
        __syncthreads();
    }
    
    // Save final result to target
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}