#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate global thread index
    uint tidx = blockDim.x * blockIdx.x + threadIdx.x;

    // Initialize the random state for each thread
    hiprand_init(seed, tidx, 0, &state[tidx]);
}