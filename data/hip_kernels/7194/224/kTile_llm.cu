#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Compute flattened index for current thread
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x; // Total threads

    // Use stride loop for coalesced memory access
    for (uint i = idx; i < tgtWidth * tgtHeight; i += stride) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}