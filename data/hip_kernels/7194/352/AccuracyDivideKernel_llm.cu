#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel function for performance
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load accuracy into register for faster access
    float acc = *accuracy;
    
    // Perform division
    acc /= N;
    
    // Store result back to global memory
    *accuracy = acc;
}