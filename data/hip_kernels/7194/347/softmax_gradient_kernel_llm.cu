#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float threadSum = 0.0f;

    // First level reduction within threads
    for (int i = idx; i < dim; i += blockDim.x) {
        threadSum += dY[i] * Y[i];
    }
    reduction_buffer[idx] = threadSum;
    __syncthreads();

    // Warp-level reduction
    if (blockDim.x >= 64) { if (idx < 32) reduction_buffer[idx] += reduction_buffer[idx + 32]; } __syncthreads();
    if (blockDim.x >= 32) { if (idx < 16) reduction_buffer[idx] += reduction_buffer[idx + 16]; } __syncthreads();
    if (blockDim.x >= 16) { if (idx < 8)  reduction_buffer[idx] += reduction_buffer[idx + 8];  } __syncthreads();
    if (blockDim.x >= 8)  { if (idx < 4)  reduction_buffer[idx] += reduction_buffer[idx + 4];  } __syncthreads();
    if (blockDim.x >= 4)  { if (idx < 2)  reduction_buffer[idx] += reduction_buffer[idx + 2];  } __syncthreads();
    if (blockDim.x >= 2)  { if (idx < 1)  reduction_buffer[idx] += reduction_buffer[idx + 1];  } __syncthreads();

    float sum = reduction_buffer[0];
    // Compute gradients
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - sum);
    }
}