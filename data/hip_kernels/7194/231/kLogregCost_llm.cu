#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = __float2int_rn(labels[tx]); // Use fast float-to-int conversion.
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        float correctProbability = 0.0f; // Initialize outside of conditions.
        if (labelp == maxp) {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp); // Avoid branching, use boolean arithmetic.
            }
            correctProbability = 1.0f / float(numMax);
        }
        correctProbs[tx] = correctProbability; // Assign once.
    }
}