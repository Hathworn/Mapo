#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposedMatrixKernel(int* d_a, int* d_b) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    // Use shared memory for better memory latency and coalesced access
    __shared__ int tile[TILE_SIZE][TILE_SIZE + 1];

    while (i < N) {
        j = threadIdx.y + blockDim.y * blockIdx.y;
        while (j < N) {
            // Load data into shared memory
            tile[threadIdx.y][threadIdx.x] = d_a[j * N + i];
            __syncthreads();

            // Write transposed data from shared memory to global memory
            d_b[i * N + j] = tile[threadIdx.x][threadIdx.y];
            __syncthreads();
            
            j += blockDim.y * gridDim.y;
        }
        i += blockDim.x * gridDim.x;
    }
}