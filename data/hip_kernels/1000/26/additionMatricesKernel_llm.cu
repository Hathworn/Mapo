#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void additionMatricesKernel(int* d_a, int* d_b, int* d_c) {
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform element-wise addition if index is within bounds
    d_c[index] = d_a[index] + d_b[index];
}