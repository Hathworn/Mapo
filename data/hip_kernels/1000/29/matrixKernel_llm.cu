#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixKernel(float* d_in, float* d_out) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Load diagonal element once to reduce memory access
    float dividend = __ldg(&d_in[by * BLOCK_SIZE * STRIDE + bx * BLOCK_SIZE]);

    // Load divisor and compute result
    float divisor = __ldg(&d_in[(by * BLOCK_SIZE + ty) * STRIDE + (bx * BLOCK_SIZE + tx)]);

    // Perform division and store the result in the output
    d_out[(by * BLOCK_SIZE + ty) * STRIDE + (bx * BLOCK_SIZE + tx)] = dividend / divisor;
}