#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposedMatrixKernel(int* d_a, int* d_b, int N) {
    // Calculate global row and column indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Transpose only if indices are within matrix bounds
    if (row < N && col < N) {
        // Perform transpose operation
        d_b[col * N + row] = d_a[row * N + col];
    }
}