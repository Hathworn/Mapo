#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void additionMatricesKernel(int* d_a, int* d_b, int* d_c) {
    // Calculate the global index for matrix access
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure we don't exceed matrix bounds and improve memory access
    if (i < N && j < N) {
        d_c[i * N + j] = d_a[i * N + j] + d_b[i * N + j];
    }
}