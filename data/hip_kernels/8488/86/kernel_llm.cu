#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *a, size_t N)
{
    // Calculate global thread ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x; 

    // Declare shared memory
    __shared__ float s[BS];

    // Calculate the number of blocks
    int blocks = (N + BS - 1) / BS;
    float sum = 0.0f;

    // Iterate over blocks to cover all elements
    for (int ib = 0; ib < blocks; ib++)
    {
        int off = ib * BS + threadIdx.x;
        
        // Load data into shared memory, check boundary
        if (off < N)
            s[threadIdx.x] = a[off];
        else
            s[threadIdx.x] = 0.0f; // Initialize with 0 if out of bounds

        __syncthreads(); // Ensure all threads have loaded the data

        // Perform reduction in shared memory
        for (int skip = BS / 2; skip > 0; skip >>= 1)
        {
            if (threadIdx.x < skip && (threadIdx.x + skip) < BS)
            {
                s[threadIdx.x] += s[threadIdx.x + skip];
            }
            __syncthreads(); // Synchronize before next step
        }

        // Accumulate result in sum
        if (threadIdx.x == 0)
        {
            sum += s[0];
        }
    }

    // Store final sum in the first element
    if (tid == 0) 
    {
        a[0] = sum;
    }
}