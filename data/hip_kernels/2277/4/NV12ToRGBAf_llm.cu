#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void YUV2RGB(uint32_t *yuvi, float *red, float *green, float *blue)
{
    const float luma = float(yuvi[0]);
    const float u    = float(yuvi[1]) - 512.0f;
    const float v    = float(yuvi[2]) - 512.0f;

    *red    = luma + 1.140f * v;
    *green  = luma - 0.395f * u - 0.581f * v;
    *blue   = luma + 2.032f * u;
}

__global__ void NV12ToRGBAf(uint32_t* srcImage, size_t nSourcePitch, float4* dstImage, size_t nDestPitch, uint32_t width, uint32_t height)
{
    int x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    uint32_t yuv101010Pel[2];
    uint32_t processingPitch = nSourcePitch;
    uint8_t* srcImageU8 = (uint8_t*)srcImage;

    yuv101010Pel[0] = (srcImageU8[y * processingPitch + x]) << 2;
    yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]) << 2;

    uint32_t chromaOffset = processingPitch * height;
    int y_chroma = y >> 1;

    if (y & 1)
    {
        uint32_t chromaCb = srcImageU8[chromaOffset + y_chroma * processingPitch + x];
        uint32_t chromaCr = srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1];

        if (y_chroma < ((height >> 1) - 1))
        {
            chromaCb = (chromaCb + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x] + 1) >> 1;
            chromaCr = (chromaCr + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x + 1] + 1) >> 1;
        }

        yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
        yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE + 2));
        yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }
    else
    {
        yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x] << (COLOR_COMPONENT_BIT_SIZE + 2));
        yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

        yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x] << (COLOR_COMPONENT_BIT_SIZE + 2));
        yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
    }

    uint32_t yuvi[6];
    float red[2], green[2], blue[2];

    yuvi[0] = (yuv101010Pel[0] & COLOR_COMPONENT_MASK);
    yuvi[1] = ((yuv101010Pel[0] >> COLOR_COMPONENT_BIT_SIZE) & COLOR_COMPONENT_MASK);
    yuvi[2] = ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    yuvi[3] = (yuv101010Pel[1] & COLOR_COMPONENT_MASK);
    yuvi[4] = ((yuv101010Pel[1] >> COLOR_COMPONENT_BIT_SIZE) & COLOR_COMPONENT_MASK);
    yuvi[5] = ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK);

    YUV2RGB(&yuvi[0], &red[0], &green[0], &blue[0]);
    YUV2RGB(&yuvi[3], &red[1], &green[1], &blue[1]);

    const float s = 1.0f / 1024.0f * 255.0f;

    dstImage[y * width + x] = make_float4(red[0] * s, green[0] * s, blue[0] * s, 1.0f);
    dstImage[y * width + x + 1] = make_float4(red[1] * s, green[1] * s, blue[1] * s, 1.0f);
}