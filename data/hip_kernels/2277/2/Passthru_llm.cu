#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define constAlpha 255

__device__ uint32_t RGBAPACK_8bit(float red, float green, float blue, uint32_t alpha)
{
    uint32_t ARGBpixel = 0;

    // Clamp final 8 bit results
    red   = min(max(red,   0.0f), 255.0f);
    green = min(max(green, 0.0f), 255.0f);
    blue  = min(max(blue,  0.0f), 255.0f);

    // Convert to 8 bit unsigned integers per color component
    ARGBpixel = ((((uint32_t)red)   << 24) |
                 (((uint32_t)green) << 16) |
                 (((uint32_t)blue)  <<  8) | alpha);

    return ARGBpixel;
}

__global__ void Passthru(uint32_t *srcImage, size_t nSourcePitch, uint32_t *dstImage, size_t nDestPitch, uint32_t width, uint32_t height)
{
    // Calculate unique thread index for processing
    int x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    uint32_t processingPitch = nSourcePitch;
    uint32_t dstImagePitch = nDestPitch >> 2;
    uint8_t *srcImageU8 = (uint8_t *)srcImage;

    // Read 2 Luma components at a time
    uint32_t yuv101010Pel[2];
    yuv101010Pel[0] = srcImageU8[y * processingPitch + x];
    yuv101010Pel[1] = srcImageU8[y * processingPitch + x + 1];

    // Extract luminance and convert to RGBA
    float luma0 = static_cast<float>(yuv101010Pel[0] & 0x00FF);
    float luma1 = static_cast<float>(yuv101010Pel[1] & 0x00FF);

    dstImage[y * dstImagePitch + x] = RGBAPACK_8bit(luma0, luma0, luma0, constAlpha);
    dstImage[y * dstImagePitch + x + 1] = RGBAPACK_8bit(luma1, luma1, luma1, constAlpha);
}