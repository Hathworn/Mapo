#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#define N 100000 // number of input values
#define R 100    // reduction factor
#define F (1+((N-1)/R)) // how many values will be in the final output

__global__ void basicRun(double *a, double *z) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= F) return; // Correct bounds check to be inclusive of F
    double avg = 0.0;
    
    // Unroll loop for performance improvement
    #pragma unroll
    for (int i = 0; i < R; i++) {
        avg += a[i + tid * R];
    }

    z[tid] = avg / R; // Divide sum by total number of input values to get average
}