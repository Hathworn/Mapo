#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce(double *a, double *z, int sizeOut) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= N / 2) return;

    extern __shared__ double subTotals[];
    // Load data into shared memory
    subTotals[threadIdx.x] = (a[tid * 2] + a[tid * 2 + 1]) / 2;
    __syncthreads();
    
    for (int level = 2; (blockDim.x / level) >= sizeOut; level *= 2) {
        if (threadIdx.x % level == 0) {
            // Reduce using fewer threads
            subTotals[threadIdx.x] = (subTotals[threadIdx.x] + subTotals[threadIdx.x + (level / 2)]) / 2;
        }
        __syncthreads(); // Synchronize after each reduction step
    }
    
    if (threadIdx.x % (level / 2) == 0) {
        // Store result in global memory
        z[tid / (level / 2)] = subTotals[threadIdx.x];
    }
}