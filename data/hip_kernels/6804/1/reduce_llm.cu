#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce(double *a, double *z, int sizeOut) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid > N / 2) return;

    extern __shared__ double subTotals[];
    // Use shared memory for reduction, double the values per thread
    subTotals[threadIdx.x] = (a[tid * 2] + a[tid * 2 + 1]) / 2.0;
    __syncthreads();

    // Optimize reduction using binary tree approach
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        if (threadIdx.x % (2 * stride) == 0) {
            subTotals[threadIdx.x] += subTotals[threadIdx.x + stride];
            subTotals[threadIdx.x] /= 2.0;
        }
        __syncthreads();
    }

    // Store the result in output array
    if (threadIdx.x == 0) {
        z[blockIdx.x] = subTotals[0];
    }
}
```
