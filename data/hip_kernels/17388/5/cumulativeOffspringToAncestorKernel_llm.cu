#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cumulativeOffspringToAncestorKernel(const int* cumulativeOffspring, int* ancestor, int numParticles) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure idx is within valid range
    if (idx >= numParticles) return;
    
    // Compute start and number of offspring for current thread
    int start = idx == 0 ? 0 : cumulativeOffspring[idx - 1];
    int numCurrentOffspring = cumulativeOffspring[idx] - start;

    // Optimize by using parallel threads to update the ancestor array
    for (int j = threadIdx.x; j < numCurrentOffspring; j += blockDim.x) {
        ancestor[start + j] = idx;
    }
}