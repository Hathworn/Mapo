#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* File smc_kernels.cu contains definitions of kernels used by SMC.
*/

#ifdef __NVCC__




#endif
__global__ void initCurandStates(hiprandState* randStates, int numThreads, int seed) {
    // Efficiently calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early exit if index is out of bounds
    if (i >= numThreads) return;
    
    // Initialize local state for better performance
    hiprandState randStateLocal;
    hiprand_init(1234 + clock64(), seed * numThreads + i, 0, &randStateLocal);
    
    // Store the local state back to global memory
    randStates[i] = randStateLocal;
}