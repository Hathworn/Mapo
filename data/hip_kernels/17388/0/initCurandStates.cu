#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* File smc_kernels.cu contains definitions of kernels used by SMC.
*/

#ifdef __NVCC__




#endif
__global__ void initCurandStates(hiprandState* randStates, int numThreads, int seed) {
int i = blockIdx.x * blockDim.x + threadIdx.x;
if(i >= numThreads || i < 0) return;

// Double check this seed, need only to be unique over one inference, as time should vary between inferences.
// hiprand_init(1234 + clock64(), seed * numThreads + i, 0, &particles->randStates[i]);
hiprandState randStateLocal = randStates[i];
hiprand_init(1234 + clock64(), seed * numThreads + i, 0, &randStateLocal);
randStates[i] = randStateLocal;
}