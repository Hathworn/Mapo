#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// __global__ function optimized with improved memory access
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) {
        float B1_t = powf(B1, t);  // Precompute B1^t
        float B2_t = powf(B2, t);  // Precompute B2^t

        float mhat = m[index] / (1.f - B1_t);  // Use precomputed B1^t
        float vhat = v[index] / (1.f - B2_t);  // Use precomputed B2^t

        x[index] += rate * mhat / (sqrtf(vhat) + eps);  // Use += for direct update
    }
}