#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    /* Use optimized calculation for unique thread index */
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    int stride = blockDim.x * gridDim.x; // Calculate stride

    // Iterate through the elements with stride for better utilization
    for (; i < N; i += stride) {
        Y[i * INCY] *= X[i * INCX];
    }
}