#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate thread ID more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we are working within the bounds of the data
    if (i < n) {
        float diff = truth[i] - pred[i];
        
        // Reduce the number of memory writes by combining operations
        float diff_squared = diff * diff;
        error[i] = diff_squared;
        delta[i] = diff;
    }
}