#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    float sum = 0;
    float largest = -INFINITY;

    // Use shared memory for reduction
    extern __shared__ float shared_data[];

    // Find the largest value
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        float val = input[i * stride];
        atomicMax((float*)&shared_data[0], val);
    }
    __syncthreads();
    largest = shared_data[0];

    // Compute exponentials and their sum
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        float e = expf((input[i * stride] - largest) / temp);
        atomicAdd(&sum, e);
        output[i * stride] = e;
    }
    __syncthreads();

    // Normalize to get softmax output
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        output[i * stride] /= sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= batch * groups) return;

    int b = id / groups;
    int g = id % groups;

    // Optimize by using block-level shared memory
    extern __shared__ float shared_data[];
    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}