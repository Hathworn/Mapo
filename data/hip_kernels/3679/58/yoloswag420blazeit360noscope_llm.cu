#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified ID calculation for 1D grid
    if(id < size) 
    {
        float rand_val = rand[id]; // Load rand memory to register
        float input_val = input[id]; // Load input memory to register
        input[id] = (rand_val < prob) ? 0 : input_val * scale; // Perform computation and store
    }
}