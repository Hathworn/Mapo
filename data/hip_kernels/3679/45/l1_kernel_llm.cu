#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved l1 kernel function
__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(i < n)
    {
        float diff = truth[i] - pred[i];
        
        // Use built-in functions for performance optimization
        error[i] = fabsf(diff);
        delta[i] = copysignf(1.0f, diff);
    }
}