#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Determine total stride for grid

    // Process elements in a grid-strided loop
    for (int idx = i; idx < n; idx += stride) {
        float t = truth[idx];
        float p = pred[idx];
        error[idx] = -t * logf(p + 1e-7f) - (1.0f - t) * logf(1.0f - p + 1e-7f); // Use logf for float precision
        delta[idx] = t - p;
    }
}