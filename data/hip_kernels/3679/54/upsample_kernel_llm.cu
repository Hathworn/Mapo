#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified 1D index calculation
    if (i >= N) return;
  
    // Calculate 3D index dimensions
    int out_w = (i % (w * stride));
    int out_h = ((i / (w * stride)) % (h * stride));
    int out_c = ((i / (w * stride * h * stride)) % c);
    int b = ((i / (w * stride * h * stride * c)) % batch);

    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_c = out_c;
    
    // Calculate 1D input index
    int in_index = b * w * h * c + in_c * w * h + in_h * w + in_w;

    // Apply forward or backward operation
    if (forward) {
        atomicAdd(&out[out_index], scale * x[in_index]); // Optimized for atomic add in forward
    } else {
        atomicAdd(x + in_index, scale * out[out_index]); // Still necessary to use atomicAdd due to possible concurrency issues
    }
}