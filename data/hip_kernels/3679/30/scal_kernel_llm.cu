#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize calculation of global thread index
    int gridSize = blockDim.x * gridDim.x;         // Calculate total number of threads in the grid
    while (i < N) {
        X[i*INCX] *= ALPHA;
        i += gridSize;                             // Advance by grid size for next iteration
    }
}