#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Use shared memory for performance improvement
    extern __shared__ float shared_weights[];
    float mean = 0;

    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[i] = fabsf(weights[f * size + i]);
    }
    __syncthreads();

    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        atomicAdd(&mean, shared_weights[i]);
    }
    __syncthreads();

    mean = mean / size;

    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }
}