#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = blockDim.x * gridDim.x;
    // Use grid-stride loop for better utilization
    while (i < (NX + NY) * B) {
        int b = i / (NX + NY);
        int j = i % (NX + NY);
        if (j < NX) {
            if (X) X[b * NX + j] += OUT[i];
        } else {
            if (Y) Y[b * NY + j - NX] += OUT[i];
        }
        i += gridStride;
    }
}