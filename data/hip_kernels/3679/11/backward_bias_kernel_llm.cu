#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Loop unrolling for performance
    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; i += BLOCK * 4) {
            int index1 = p + i + size * (filter + n * b);
            int index2 = index1 + BLOCK;
            int index3 = index2 + BLOCK;
            int index4 = index3 + BLOCK;

            sum += (p + i < size) ? delta[index1] : 0;
            sum += (p + i + BLOCK < size) ? delta[index2] : 0;
            sum += (p + i + 2 * BLOCK < size) ? delta[index3] : 0;
            sum += (p + i + 3 * BLOCK < size) ? delta[index4] : 0;
        }
    }

    part[p] = sum;
    __syncthreads();

    // Reduction using a single thread
    if (p == 0) {
        for (int i = 0; i < BLOCK; ++i) {
            bias_updates[filter] += part[i];
        }
    }
}