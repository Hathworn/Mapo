#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = blockIdx.x * blockDim.x + threadIdx.x;
    if (s >= size) return;
    
    // Use shared memory to reduce global memory accesses
    extern __shared__ float shared_input[];
    
    // Load data into shared memory
    for (int i = 0; i < n; ++i) {
        shared_input[i] = fabsf(input[i * size + s]);
    }
    
    // Calculate mean
    float mean = 0;
    for (int i = 0; i < n; ++i) {
        mean += shared_input[i];
    }
    mean /= n;
    
    // Binarize input
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}