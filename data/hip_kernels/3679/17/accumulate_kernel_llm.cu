#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Calculate global index
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;
    
    float local_sum = 0;  // Use local variable for accumulation to reduce global memory access
    for (int k = 0; k < n; ++k) {
        local_sum += x[k * groups + i];
    }
    sum[i] = local_sum;  // Write the result once outside the loop
}