#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid-stride loop for better scalability
    for (int stride = i; stride < N; stride += blockDim.x * gridDim.x)
    {
        X[stride * INCX] = ALPHA;
    }
}