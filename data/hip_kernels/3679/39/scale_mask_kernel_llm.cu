#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    // Compute global index with 1D grid structure for improved calculation efficiency
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a while loop for grid-stride approach
    while (i < n) {
        if (mask[i] == mask_num) {
            x[i] *= scale; // Perform scaling if condition is met
        }
        i += blockDim.x * gridDim.x; // Move to the next position in the grid-stride loop
    }
}