#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    // Calculate indices
    int k = id % c;
    int b = id / c;

    // Precompute factor to minimize redundant division
    float scale = 1.0f / (w * h);
    
    // Optimize loop by using unrolled access pattern for in_delta
    for(int i = 0; i < w * h; ++i){
        // Compute indices once per iteration
        int in_index = i + h * w * (k + b * c);
        int out_index = k + c * b;

        // Update in_delta using precomputed scale
        in_delta[in_index] += out_delta[out_index] * scale;
    }
}