#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float s1, float s2, float *out)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return;

    // Pre-compute reused indices to enhance performance
    int i = (id % minw) * sample; 
    id /= minw;
    int j = (id % minh) * sample; 
    id /= minh;
    int k = id % minc;
    id /= minc;
    int b = id % batch;

    // Use pre-computed indices for address calculations
    int out_index = i + w2 * (j + h2 * (k + c2 * b));
    int add_index = (i / sample) * stride + w1 * ((j / sample) * stride + h1 * (k + c1 * b));
    
    // Perform operation using new indices
    out[out_index] = s1 * out[out_index] + s2 * add[add_index];
}