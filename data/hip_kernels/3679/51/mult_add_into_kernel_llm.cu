#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Use a 1D grid and block calculation for improved efficiency
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n){
        // Perform the operation only for valid indices
        c[i] += a[i] * b[i];
    }
}