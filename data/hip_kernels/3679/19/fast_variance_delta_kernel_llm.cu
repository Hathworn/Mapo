#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Initialize shared memory
    local[id] = 0;

    // Loop unrolling - process multiple elements per thread to improve efficiency
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            if (i < spatial) {
                local[id] += delta[index] * (x[index] - mean[filter]);
            }
        }
    }

    // Sum reduction within block
    __syncthreads();
    
    if (threads >= 1024) { if (id < 512) { local[id] += local[id + 512]; } __syncthreads(); }
    if (threads >= 512) { if (id < 256) { local[id] += local[id + 256]; } __syncthreads(); }
    if (threads >= 256) { if (id < 128) { local[id] += local[id + 128]; } __syncthreads(); }
    if (threads >= 128) { if (id < 64) { local[id] += local[id + 64]; } __syncthreads(); }
    
    if (id < 32) {
        volatile float* vlocal = local; // Avoid register spill
        if (threads >= 64) vlocal[id] += vlocal[id + 32];
        if (threads >= 32) vlocal[id] += vlocal[id + 16];
        if (threads >= 16) vlocal[id] += vlocal[id + 8];
        if (threads >= 8) vlocal[id] += vlocal[id + 4];
        if (threads >= 4) vlocal[id] += vlocal[id + 2];
        if (threads >= 2) vlocal[id] += vlocal[id + 1];
    }

    // Final result stored by first thread
    if (id == 0) {
        variance_delta[filter] = local[0] * -.5f * powf(variance[filter] + .00001f, -1.5f);
    }
}