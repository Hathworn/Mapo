#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel using shared memory for parallel computations
__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Compute the linear index of a thread within a block

    if(i < N) {
        // Load value into a register for quick access
        float xi = X[i * INCX];

        // Apply constraints using fminf and fmaxf functions
        X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, xi));
    }
}