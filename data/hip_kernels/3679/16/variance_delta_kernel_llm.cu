#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float var_d = 0.0f;
    float mean_val = mean[i];
    float variance_val = variance[i];
    float factor = -.5f * powf(variance_val + .00001f, -1.5f);

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            var_d += delta[index] * (x[index] - mean_val);
        }
    }
    variance_delta[i] = var_d * factor; // Pre-compute reusable values and optimize memory operations
}