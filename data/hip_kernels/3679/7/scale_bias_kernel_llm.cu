#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Improved performance by ensuring that memory access is coalesced
    if (offset < size) {
        int index = (batch * n + filter) * size + offset;
        output[index] *= biases[filter];
    }
}