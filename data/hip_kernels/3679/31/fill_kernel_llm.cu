#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Improved indexing for better memory access
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Fuse condition to avoid branching
    X[i * INCX] = ALPHA * (i < N);
}