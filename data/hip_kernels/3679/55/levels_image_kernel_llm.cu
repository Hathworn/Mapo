#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float3 hsv_to_rgb_kernel(float3 hsv) {
    float h = hsv.x;
    float s = hsv.y;
    float v = hsv.z;

    float r, g, b;
    float f, p, q, t;

    if (s == 0) {
        r = g = b = v;
    } else {
        int index = (int) floorf(h);
        f = h - index;
        p = v * (1 - s);
        q = v * (1 - s * f);
        t = v * (1 - s * (1 - f));
        
        // Use lookup to avoid multiple if-statements
        switch (index) {
            case 0: r = v; g = t; b = p; break;
            case 1: r = q; g = v; b = p; break;
            case 2: r = p; g = v; b = t; break;
            case 3: r = p; g = q; b = v; break;
            case 4: r = t; g = p; b = v; break;
            default: r = v; g = p; b = q; break;
        }
    }

    r = fminf(fmaxf(r, 0.0f), 1.0f);
    g = fminf(fmaxf(g, 0.0f), 1.0f);
    b = fminf(fmaxf(b, 0.0f), 1.0f);
    
    return make_float3(r, g, b);
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb) {
    float r = rgb.x, g = rgb.y, b = rgb.z;
    float h, s, v;
    
    float max = fmaxf(r, fmaxf(g, b));
    float min = fminf(r, fminf(g, b));
    float delta = max - min;
    
    v = max;
    if (max == 0) {
        s = 0;
        h = -1;
    } else {
        s = delta / max;
        if (r == max) {
            h = (g - b) / delta;
        } else if (g == max) {
            h = 2 + (b - r) / delta;
        } else {
            h = 4 + (r - g) / delta;
        }
        if (h < 0) h += 6;
    }

    return make_float3(h, s, v);
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift) {
    int size = batch * w * h;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return;
    
    int x = id % w;
    id /= w;
    int y = id % h;
    id /= h;

    // Precompute indices and parameters
    size_t offset = id * h * w * 3;
    float rshift = rand[0];
    float gshift = rand[1];
    float bshift = rand[2];
    float r0 = rand[8 * id];
    float r1 = rand[8 * id + 1];
    float r2 = rand[8 * id + 2];
    float r3 = rand[8 * id + 3];

    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > 0.5f) ? 1.f / saturation : saturation;
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > 0.5f) ? 1.f / exposure : exposure;

    image += offset;
    float r = image[x + w * (y + h * 0)];
    float g = image[x + w * (y + h * 1)];
    float b = image[x + w * (y + h * 2)];
    float3 rgb = make_float3(r, g, b);
    
    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }
    
    // Efficiently write back the computed values
    image[x + w * (y + h * 0)] = rgb.x * scale + translate + (rshift - 0.5f) * shift;
    image[x + w * (y + h * 1)] = rgb.y * scale + translate + (gshift - 0.5f) * shift;
    image[x + w * (y + h * 2)] = rgb.z * scale + translate + (bshift - 0.5f) * shift;
}