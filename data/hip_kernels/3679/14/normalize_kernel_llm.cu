#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (index >= N) return;
    
    // Calculate filter index for the current element
    int f = (index / spatial) % filters;

    // Normalize the element using pre-computed mean and variance
    float norm = sqrtf(variance[f] + 1e-5f);
    x[index] = (x[index] - mean[f]) / norm;
}