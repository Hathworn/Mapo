#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Optimize thread index calculation using grid-stride loop
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x)
    {
        X[i * INCX] += ALPHA;
    }
}