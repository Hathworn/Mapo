#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Exit if index is beyond N
    if (i >= N) return;

    int in_index = i;
    int in_w = i % w;
    i /= w;  // Simplified division
    int in_h = i % h;
    i /= h;
    int in_c = i % c;
    int b = i / c;  // Simplified division to avoid using additional divide
    
    // Precompute out channel
    int out_c = c / (stride * stride);

    // Precompute offsets
    int offset = in_c / out_c;
    int c2 = in_c % out_c;
    int w2 = in_w * stride + offset % stride;  
    int h2 = in_h * stride + offset / stride;  
    
    // Calculate output index
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Conditional assignment to output
    if (forward) 
        out[out_index] = x[in_index];
    else 
        out[in_index] = x[out_index];
}