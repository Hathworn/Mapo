#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;

    // Find largest element
    for(i = 0; i < n; ++i){
        int val = input[i*stride];
        largest = (val>largest) ? val : largest;
    }

    // Compute exponential values and accumulate sum
    for(i = 0; i < n; ++i){
        float e = expf(input[i*stride]/temp - largest/temp);
        sum += e;
        output[i*stride] = e;
    }

    // Normalize output
    for(i = 0; i < n; ++i){
        output[i*stride] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Optimize ID calculation

    if (id >= spatial*batch*groups) return;

    int s = id % spatial;
    int b = (id / spatial) / groups; // Optimize batch and group calculation
    int g = (id / spatial) % groups; 

    int goff = group_offset[g] * spatial;
    int boff = b * stride;

    // Call optimized device function
    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}