#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;
    
    // Precompute base indices for efficiency
    for (int b = 0; b < batch; ++b) {
        int base1 = b * size * n + f1 * size;
        int base2 = b * size * n + f2 * size;
        for (int i = 0; i < size; ++i) {
            int i1 = base1 + i;
            int i2 = base2 + i;
            float val1 = output[i1];
            float val2 = output[i2];
            sum += val1 * val2;
            norm1 += val1 * val1;
            norm2 += val2 * val2;
        }
    }

    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    sum = sum / norm;
    
    // Reuse precomputed base indices here as well
    for (int b = 0; b < batch; ++b) {
        int base1 = b * size * n + f1 * size;
        int base2 = b * size * n + f2 * size;
        for (int i = 0; i < size; ++i) {
            int i1 = base1 + i;
            int i2 = base2 + i;
            delta[i1] += -scale * sum * output[i2] / norm;
            delta[i2] += -scale * sum * output[i1] / norm;
        }
    }
}