#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    float scale = 1.f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Combine blockIdx.y and gridDim.x with blockIdx.x for 1D grid
    if (i >= filters) return;
    
    float sum = 0.0f; // Use local variable for accumulation
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += x[index]; // Accumulate in local memory
        }
    }
    mean[i] = sum * scale; // Set the mean after accumulation
}