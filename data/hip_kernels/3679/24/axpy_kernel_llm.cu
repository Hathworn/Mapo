#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use loop to increase occupancy and handle larger data sizes
    for (; i < N; i += stride) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}