#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f; // Initialize sum as float

    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) { // Optimize loop to reduce divergence
            int index = i + size * (filter + n * b);
            sum += delta[index] * x_norm[index];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Use parallel reduction for less contention
    for (int stride = BLOCK / 2; stride > 0; stride /= 2) {
        if (p < stride) {
            part[p] += part[p + stride];
        }
        __syncthreads();
    }

    // Atomically update the scale_updates array
    if (p == 0) {
        atomicAdd(&scale_updates[filter], part[0]);
    }
}