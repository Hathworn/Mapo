#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Check index within bounds and mask condition
    if (i < n && mask[i] == mask_num) {
        x[i] = val;
    }
}