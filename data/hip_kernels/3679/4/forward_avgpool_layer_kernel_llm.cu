#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Simplified 1D grid launch
    if(id >= n * c) return; // Should iterate over n * c

    int k = id % c; 
    int b = id / c; 

    int out_index = k + c * b;
    float sum = 0.0f;
    int offset = h * w * (k + b * c);
    
    for(int i = 0; i < w * h; ++i){
        sum += input[i + offset];
    }

    output[out_index] = sum / (w * h); // Pre-divide to improve performance
}