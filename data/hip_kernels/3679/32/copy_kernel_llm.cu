#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Optimize thread index calculation for 1D grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use memory coalescing for better performance
    if (i < N) 
    {
        int idxX = i * INCX + OFFX;
        int idxY = i * INCY + OFFY;
        Y[idxY] = X[idxX];
    }
}