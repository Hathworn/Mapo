#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Combine inner loops to minimize index calculations
    for(int j = 0; j < batch; ++j){
        for(int i = id; i < spatial; i += threads){
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += delta[index];
        }
    }

    __syncthreads();

    // Optimize reduction with stride loop
    if (id == 0) {
        for (int stride = 1; stride < threads; stride *= 2) {
            if (id + stride < threads) {
                local[id] += local[id + stride];
            }
            __syncthreads();
        }

        mean_delta[filter] = local[0] * (-1.f / sqrtf(variance[filter] + .00001f));
    }
}