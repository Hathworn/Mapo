#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Return early if out of filter index
    if (i >= filters) return;

    // Use shared memory to reduce global memory access
    __shared__ float shared_delta;
    shared_delta = 0.0f;

    int j, k;
    float local_mean_delta = 0.0f;
    for (j = 0; j < batch; j++) {
        for (k = 0; k < spatial; k++) {
            int index = j * filters * spatial + i * spatial + k;
            local_mean_delta += delta[index];
        }
    }
    // Normalize mean_delta using variance
    mean_delta[i] = local_mean_delta * (-1.f / sqrtf(variance[i] + .00001f));
}