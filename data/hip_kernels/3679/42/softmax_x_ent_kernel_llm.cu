#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds
    if(i < n) {
        float p = pred[i];
        // Use ternary operation for concise conditional computation
        error[i] = (truth[i]) ? -logf(p) : 0.0f;
        delta[i] = truth[i] - p;
    }
}