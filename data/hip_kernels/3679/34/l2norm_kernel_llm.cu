#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    // Calculate index for the current thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    // Determine the batch and spatial indices
    int b = index / spatial;
    int i = index % spatial;
    float sum = 0.0f;
    int baseIndex = b * filters * spatial + i;

    // Unroll the loop to optimize memory access
    for (int f = 0; f < filters; ++f) {
        int elemIndex = baseIndex + f * spatial;
        float val = x[elemIndex];
        sum += val * val;
    }

    // Compute the norm and avoid division by zero
    sum = sqrtf(sum);
    if (sum == 0.0f) sum = 1.0f;

    // Update x and dx arrays
    for (int f = 0; f < filters; ++f) {
        int elemIndex = baseIndex + f * spatial;
        x[elemIndex] /= sum;
        dx[elemIndex] = (1.0f - x[elemIndex]) / sum;
    }
}