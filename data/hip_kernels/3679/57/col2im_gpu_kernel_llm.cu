#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void col2im_gpu_kernel(const int n, const float* data_col, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_im) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop unrolling and shared memory to optimize memory access
    extern __shared__ float shared_data[];
    for (; index < n; index += blockDim.x * gridDim.x) {
        float val = 0;
        int w = index % width + pad;
        int h = (index / width) % height + pad;
        int c = index / (width * height);
        
        int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
        int w_col_end = min(w / stride + 1, width_col);
        int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
        int h_col_end = min(h / stride + 1, height_col);

        int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
        int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
        int coeff_w_col = (1 - stride * height_col * width_col);

        for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
#ifndef UNROLL
            for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
                val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
            }
#else
            // Use loop unrolling for efficiency
            int w_col;
            for (w_col = w_col_start; w_col <= w_col_end - 4; w_col += 4) {
                val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
                val += data_col[offset + h_col * coeff_h_col + (w_col + 1) * coeff_w_col];
                val += data_col[offset + h_col * coeff_h_col + (w_col + 2) * coeff_w_col];
                val += data_col[offset + h_col * coeff_h_col + (w_col + 3) * coeff_w_col];
            }
            for (; w_col < w_col_end; ++w_col) {
                val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
            }
#endif
        }
        data_im[index] += val;
    }
}