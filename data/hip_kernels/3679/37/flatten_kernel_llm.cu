#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate the flattened index of the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a more efficient loop boundary check
    if (i >= N) return;
    
    // Calculate spatial, channel, and batch indices using division and modulo
    int in_s = i % spatial;
    int in_c = (i / spatial) % layers;
    int b = i / (spatial * layers);
    
    // Compute positions based on forward flag
    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers +  in_c;
    
    // Conditional assignment based on forward flag
    if (forward) {
        out[i2] = x[i1];
    } else {
        out[i1] = x[i2];
    }
}