#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Calculate scale outside the loop
    float scale = 1.f / (batch * spatial - 1);
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= filters) return;

    float var = 0;  // Use local variable for accumulation
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + tid * spatial + k;
            float diff = x[index] - mean[tid];
            var += diff * diff; // Avoid using powf for better performance
        }
    }
    variance[tid] = var * scale; // Write to global memory outside the loop
}