#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    // Calculate the output dimensions
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int area = (size - 1) / stride;
    int pad_half = pad / 2;

    // Compute unique thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Determine position in input tensor
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    // Initialize accumulation variable
    float d = 0;

    // Use loop unrolling for optimization
    #pragma unroll
    for (int l = -area; l <= area; ++l) {
        int out_h = (i - pad_half) / stride + l;
        if (out_h < 0 || out_h >= h) continue;

        #pragma unroll
        for (int m = -area; m <= area; ++m) {
            int out_w = (j - pad_half) / stride + m;
            if (out_w < 0 || out_w >= w) continue;

            // Compute output index
            int out_index = out_w + w * (out_h + h * (k + in_c * b));

            // Accumulate delta where index matches
            if (indexes[out_index] == id) {
                d += delta[out_index];
            }
        }
    }

    // Update prev_delta with accumulated values
    prev_delta[b * in_c * in_h * in_w + k * in_h * in_w + i * in_w + j] += d;
}