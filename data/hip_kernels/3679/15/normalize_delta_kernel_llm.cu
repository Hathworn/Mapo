#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    // Calculate the global thread index to avoid redundant calculations.
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds for kernel execution.
    if (index < N) {
        // Calculate filter index using modulo operation.
        int f = (index / spatial) % filters;

        // Precompute reused expressions to enhance performance.
        float inv_sqrt_variance = rsqrtf(variance[f] + 0.00001f); // Use rsqrtf for better precision and speed
        float variance_delta_coeff = variance_delta[f] * 2.f / (spatial * batch);
        float mean_delta_coeff = mean_delta[f] / (spatial * batch);

        // Update delta with optimized expression.
        delta[index] = delta[index] * inv_sqrt_variance + variance_delta_coeff * (x[index] - mean[f]) + mean_delta_coeff;
    }
}