#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop over the large data in steps to optimize memory access
    for (; i < N; i += stride) {
        Y[i * INCY] = powf(X[i * INCX], ALPHA); // Use powf for single-precision float
    }
}