#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    extern __shared__ float local[];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0;
    int i, j;

    // Calculate sum of items handled by this thread
    for (j = 0; j < batch; ++j) {
        for (i = id; i < spatial; i += blockDim.x) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += x[index];
        }
    }

    __syncthreads();

    // Reduce within block
    if (id == 0) {
        float block_sum = 0;
        for (i = 0; i < blockDim.x; ++i) {
            block_sum += local[i];
        }
        mean[filter] = block_sum / (spatial * batch);
    }
}