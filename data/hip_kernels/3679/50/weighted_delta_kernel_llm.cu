#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified grid index calculation
    int stride = blockDim.x * gridDim.x;           // Calculate stride for grid-stride loop

    for (; i < n; i += stride) {                   // Grid-stride loop for better occupancy
        float dc_val = dc[i];                      // Load dc[i] once for efficiency
        float s_val = s[i];                        // Load s[i] once for efficiency
        if(da) da[i] += dc_val * s_val;            // Update da with preloaded values
        if(db) db[i] += dc_val * (1 - s_val);      // Update db with preloaded values
        ds[i] += dc_val * (a[i] - b[i]);           // Update ds with preloaded values
    }
}