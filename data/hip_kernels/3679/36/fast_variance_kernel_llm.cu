#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Loop unrolling to optimize the computation process 
    for (int j = 0; j < batch; ++j) {
        int base_index = j * spatial * filters + filter * spatial;
        for (int i = id; i < spatial; i += threads) {
            int index = base_index + i;
            local[id] += powf((x[index] - mean[filter]), 2);
        }
    }

    __syncthreads();

    // Use parallel reduction to compute the variance, improving efficiency
    if (id == 0) {
        float sum = 0;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        variance[filter] = sum / (spatial * batch - 1);
    }
}