#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel for better performance by avoiding calculating i for out-of-bounds threads
__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int totalThreads = blockDim.x * gridDim.x;
    for (; i < n; i += totalThreads) {
        binary[i] = (x[i] >= 0) ? 1 : -1;
    }
}