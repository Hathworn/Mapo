#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    // Calculate the global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) return;

    float sum = 0;
    // Unroll loop for better performance
    for (int b = 0; b < batch; ++b) {
        int i = b * n + index;
        sum += delta[i];
    }
    // Use atomic operation to minimize write conflicts 
    atomicAdd(&bias_updates[index], sum);
}