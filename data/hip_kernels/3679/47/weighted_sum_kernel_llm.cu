#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Optimize index calculation for 1D grid
    if(i < n)
    {
        float b_value = b ? b[i] : 0;               // Store conditional value to avoid repeated logic
        c[i] = s[i] * a[i] + (1 - s[i]) * b_value;  // Use stored value
    }
}