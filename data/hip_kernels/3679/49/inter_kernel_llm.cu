#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate the unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Guard for the total number of output elements
    if(i < (NX + NY) * B)
    {
        // Calculate batch index and element index
        int b = i / (NX + NY);
        int j = i % (NX + NY);

        // Use ternary operator for simplified condition
        OUT[i] = (j < NX) ? X[b * NX + j] : Y[b * NY + j - NX];
    }
}