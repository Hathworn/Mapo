#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Optimize using blockIdx.x as the only thread performs the calculation
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        *accuracy /= N;  // Division operation
    }
}