#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    // Initialize shared memory
    reduction_buffer[idx] = 0;
    __syncthreads(); // Ensure shared memory is ready

    float tmp = 0;
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Single thread reduces values within shared memory
    if (idx == 0) {
        tmp = 0;
        for (int i = 0; i < blockDim.x; ++i) {
            tmp += reduction_buffer[i];
        }
        reduction_buffer[0] = tmp;
    }
    __syncthreads();

    // Compute gradient
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}