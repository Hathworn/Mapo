#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//#define DEPTH 2

__device__ float dp_criteria(float *dp, int ind, int depth_dim_size, int d, float P_one, float P_two, float *d_zero, float *d_one, float *d_two, float *d_three) {
    *d_zero = dp[ind];
    *d_one = (d > 0) ? dp[ind - depth_dim_size] + P_one : 10000000;
    *d_two = (d < D - 1) ? dp[ind + depth_dim_size] + P_one : 10000000;
    return fminf(fminf(*d_zero, *d_one), fminf(*d_two, *d_three)) - *d_three + P_two;
}

__global__ void __diagonal_tl_br_aggregate(float *dp, float *cost_image, int m, int n) {
    int start_col = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int depth_dim_size = m * n;

    while (start_col < n) {
        int col = start_col;
        for (int row = 1; row < m; row++) {
            float prev_min = 100000000.0f;
            int ind = (row - 1) * n + col - 1;

            // Calculate minimum disparity cost for this column from row-1
            for (int depth = 0; depth < D; depth += D_STEP) {
                prev_min = fminf(dp[ind], prev_min);
                ind += (depth_dim_size * D_STEP);
            }

            float d0 = 0, d1 = 0, d2 = 0, d3 = prev_min + (float)P2;
            ind = (row - 1) * n + col - 1;
            int current_ind = row * n + col;

            // Unroll loop to improve performance
            #pragma unroll
            for (int d = 0; d < D; d += D_STEP) {
                dp[current_ind] += cost_image[current_ind] + dp_criteria(dp, ind, depth_dim_size, d, (float)P1, (float)P2, &d0, &d1, &d2, &d3);
                ind += (depth_dim_size * D_STEP);
                current_ind += (depth_dim_size * D_STEP);
            }

            col += 1;
            if (col == n) col = 1; // Wrap each thread around once it gets to the last column
        }
        start_col += blockDim.x;
    }
}