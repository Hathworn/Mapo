#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float dp_criteria(float *dp, int ind, int depth_dim_size, int d, float P_one, float P_two, float *d_zero, float *d_one, float *d_two, float *d_three) {
    *d_zero = dp[ind];
    *d_one = (d > 0) ? dp[ind - depth_dim_size] + P_one : 10000000;
    *d_two = (d < D-1) ? dp[ind + depth_dim_size] + P_one : 10000000;
    return fminf(fminf(*d_zero, *d_one), fminf(*d_two, *d_three)) - *d_three + P_two;
}

__global__ void __vertical_aggregate_down(float *dp, float *cost_image, int m, int n) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int depth_dim_size = m * n;

    while(col < n) {
        for (int row = 1; row < m; row++) {
            float prev_min = 100000000.0;
            int ind = (row - 1) * n + col;

            // Unroll loop explicitly to improve performance
            #pragma unroll
            for (int depth = 0; depth < D; depth += D_STEP) {
                prev_min = fminf(dp[ind], prev_min);
                ind += (depth_dim_size * D_STEP);
            }

            float d0 = 0, d1 = 0, d2 = 0;
            float d3 = prev_min + (float) P2;
            ind = (row - 1) * n + col;
            int current_ind = row * n + col;

            // Unroll loop explicitly to improve performance
            #pragma unroll
            for (int d = 0; d < D; d += D_STEP) {
                dp[current_ind] += cost_image[current_ind] + dp_criteria(dp, ind, depth_dim_size, d, (float) P1, (float) P2, &d0, &d1, &d2, &d3);
                ind += (depth_dim_size * D_STEP);
                current_ind += (depth_dim_size * D_STEP);
            }
        }
        col += gridDim.x * blockDim.x; // Advance column calculation by entire grid
    }
}