#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//#define DEPTH 2

// Optimized kernel for 3D matrix argmin computation
__global__ void argmin_3d_mat(float *dp, int *stereo_im, int m, int n)
{
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int imsize = m * n;
    int loop_limit = D * imsize;

    if (col < n && row < m) { // Use boundary checks inside loop
        int min_ind = -1;
        float current_min = 100000000.0f;
        int current_val = row * n + col;
        int v = 0;

        for (int depth = 0; depth < loop_limit; depth += imsize) {
            float val = dp[depth + current_val]; // Cache the value
            if (val < current_min) {
                min_ind = v;
                current_min = val;
            }
            v++;
        }
        stereo_im[current_val] = min_ind;
    }
}