#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//#define DEPTH 2

__device__ float dp_criteria(float *dp, int ind, int depth_dim_size, int d, float P_one, float P_two, float * d_zero, float * d_one, float * d_two, float * d_three){
    *d_zero = dp[ind];
    if (d > 0)
        *d_one = dp[ind - depth_dim_size] + P_one;
    else
        *d_one = 10000000;

    if (d < D-1)
        *d_two = dp[ind + depth_dim_size] + P_one;
    else
        *d_two = 10000000;
    
    return fminf(fminf(*d_zero, *d_one), fminf(*d_two, *d_three)) - *d_three + P_two;
}

__global__ void __diagonal_br_tl_aggregate(float *dp, float *cost_image, int m, int n)
{
    int start_col = (n - 2) - (blockDim.x * blockIdx.x) - threadIdx.x;
    int depth_dim_size = m * n;

    // Optimize loop with early out to reduce unnecessary calculations
    while(start_col >= 0)
    {
        int col = start_col;
        for (int row = m - 2; row >= 0; row--)
        {
            float prev_min = 100000000.0f;
            int ind = (row + 1) * n + col + 1;

            // Unroll the depth loop to increase efficiency
            #pragma unroll
            for (int depth = 0; depth < D; depth += D_STEP){
                prev_min = fminf(dp[ind], prev_min);
                ind += (depth_dim_size * D_STEP);
            }

            float d0, d1, d2, d3 = prev_min + (float) P2;
            ind = (row + 1) * n + col + 1;
            int current_ind = row * n + col;

            // Optimize the d loop using fewer conditions
            #pragma unroll
            for (int d = 0; d < D; d += D_STEP){
                dp[current_ind] += cost_image[current_ind] + dp_criteria(dp, ind, depth_dim_size, d, (float) P1, (float) P2, &d0, &d1, &d2, &d3);
                ind += (depth_dim_size * D_STEP);
                current_ind += (depth_dim_size * D_STEP);
            }

            col -= 1;
            if (col < 0)
                col = n - 2;
        }
        start_col -= blockDim.x;
    }
}