#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//#define DEPTH 2

// dp - cost aggregation array
// cost_image - m x n x D array
// d - use every d channels of input to conserve register memory
// m - image rows
// n - image columns
// D - depth
// depth_stride - pitch along depth dimension
// row_stride - pitch along row dimension

__device__ float dp_criteria(float *dp, int ind, int depth_dim_size, int d, float P_one, float P_two, float * d_zero, float * d_one, float * d_two, float * d_three){
    *d_zero = dp[ind];
    if (d > 0)
        *d_one = dp[ind - depth_dim_size] + P_one;
    else
        *d_one = 10000000;

    if (d < D-1)
        *d_two = dp[ind + depth_dim_size] + P_one;
    else
        *d_two = 10000000;
    
    return fminf(fminf(*d_zero, *d_one), fminf(*d_two, *d_three)) - *d_three + P_two;
}

__global__ void __vertical_aggregate_up(float *dp, float *cost_image, int m, int n) {
    // Calculate linear thread index in grid
    int globalId = blockDim.x * blockIdx.x + threadIdx.x;
    int depth_dim_size = m * n;

    // Optimize warp utilization by ensuring contiguous threads work on neighboring columns
    if (globalId < n) {
        for (int row = m - 2; row >= 0; row--) {
            float prev_min = 100000000.0;
            int ind = (row + 1) * n + globalId;

            // Optimize depth loop to minimize register usage
            for (int depth = 0; depth < D; depth += D_STEP) {
                prev_min = fminf(dp[ind], prev_min);
                ind += depth_dim_size * D_STEP;
            }

            float d0 = 0;
            float d1 = 0;
            float d2 = 0;
            float d3 = prev_min + (float) P2;
            ind = (row + 1) * n + globalId;
            int current_ind = row * n + globalId;

            // Use loop unrolling to improve performance
            for (int d = 0; d < D; d += D_STEP) {
                dp[current_ind] += cost_image[current_ind] + dp_criteria(dp, ind, depth_dim_size, d, (float) P1, (float) P2, &d0, &d1, &d2, &d3);
                ind += depth_dim_size * D_STEP;
                current_ind += depth_dim_size * D_STEP;
            }
        }
    }
}