#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//#define DEPTH 2

__device__ float dp_criteria(float *dp, int ind, int depth_dim_size, int d, float P_one, float P_two, float *d_zero, float *d_one, float *d_two, float *d_three){
    // Unrolling the if-else to ensure consistent operation within specific range
    *d_zero = dp[ind];
    *d_one = (d > 0) ? dp[ind - depth_dim_size] + P_one : 10000000;
    *d_two = (d < D-1) ? dp[ind + depth_dim_size] + P_one : 10000000;
    return fminf(fminf(*d_zero, *d_one), fminf(*d_two, *d_three)) - *d_three + P_two;
}

__global__ void __diagonal_bl_tr_aggregate(float *dp, float *cost_image, int m, int n)
{
    int start_col = blockDim.x * blockIdx.x + threadIdx.x + 1;
    int depth_dim_size = m * n;

    // Load previous minimum value into a register at the start of the row
    while (start_col < n)
    {
        int col = start_col;
        for (int row = m - 2; row >= 0; row--)
        {
            float prev_min = 100000000.0;
            int ind = (row + 1) * n + col - 1;

            // Pre-compute starting index and reuse it efficiently
            for (int depth = 0; depth < D; depth += D_STEP) {
                prev_min = fminf(dp[ind], prev_min);
                ind += depth_dim_size * D_STEP;
            }

            float d0 = 0, d1 = 0, d2 = 0, d3 = prev_min + (float) P2;
            ind = (row + 1) * n + col - 1;
            int current_ind = row * n + col;

            // Unrolling and restructuring loop for better performance
            for (int d = 0; d < D; d += D_STEP) {
                dp[current_ind] += cost_image[current_ind] + dp_criteria(dp, ind, depth_dim_size, d, (float) P1, (float) P2, &d0, &d1, &d2, &d3);
                ind += depth_dim_size * D_STEP;
                current_ind += depth_dim_size * D_STEP;
            }

            col += 1;
            if (col == n)
                col = 1;
        }
        start_col += blockDim.x;
    }
}