#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//#define DEPTH 2

__device__ float dp_criteria(float *dp, int ind, int depth_dim_size, int d, float P_one, float P_two, float *d_zero, float *d_one, float *d_two, float *d_three) {
    *d_zero = dp[ind];
    if (d > 0)
        *d_one = dp[ind - depth_dim_size] + P_one;
    else
        *d_one = 10000000;

    if (d < DEPTH-1)
        *d_two = dp[ind + depth_dim_size] + P_one;
    else
        *d_two = 10000000;
    
    return fminf(fminf(*d_zero, *d_one), fminf(*d_two, *d_three)) - *d_three + P_two;
}

__global__ void __diagonal_tr_bl_aggregate(float *dp, float *cost_image, int m, int n) {
    int depth_dim_size = m * n;
    int start_col = n - 2 - blockDim.x * blockIdx.x - threadIdx.x;

    while (start_col >= 0) {
        int col = start_col;
        for (int row = 1; row < m; ++row) {
            float prev_min = 100000000.0;
            int ind = (row - 1) * n + col + 1;

            // Min cost disparity computation for previous row
            for (int depth = 0; depth < DEPTH; depth += D_STEP) {
                prev_min = fminf(dp[ind], prev_min);
                ind += depth_dim_size * D_STEP;
            }

            float d0, d1, d2, d3 = prev_min + P2;
            ind = (row - 1) * n + col + 1;
            int current_ind = row * n + col;

            // Cost aggregation loop for disparities
            for (int d = 0; d < DEPTH; d += D_STEP) {
                dp[current_ind] += cost_image[current_ind] + dp_criteria(dp, ind, depth_dim_size, d, P1, P2, &d0, &d1, &d2, &d3);
                ind += depth_dim_size * D_STEP;
                current_ind += depth_dim_size * D_STEP;
            }

            col -= 1;
            if (col < 0)
                col = n - 2; // Wrap around column
        }
        start_col -= blockDim.x;
    }
}