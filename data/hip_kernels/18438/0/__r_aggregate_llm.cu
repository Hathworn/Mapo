#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
//#define DEPTH 2

__device__ float dp_criteria(float *dp, int ind, int depth_dim_size, int d, float P_one, float P_two, float *d_zero, float *d_one, float *d_two, float *d_three) {
    // Optimize: Inline conditional checks with ternary operators
    *d_zero = dp[ind];
    *d_one = (d > 0) ? dp[ind - depth_dim_size] + P_one : 10000000;
    *d_two = (d < D-1) ? dp[ind + depth_dim_size] + P_one : 10000000;
    
    return fminf(fminf(*d_zero, *d_one), fminf(*d_two, *d_three)) - *d_three + P_two;
}

__global__ void __r_aggregate(float *dp, float *cost_image, int m, int n) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x;
    int depth_dim_size = m * n;

    __shared__ float MinArray[SHMEM_SIZE][SHMEM_SIZE];
    int K = 0;

    while ((col < n) & (row < m)) {
        int ind = row * n + col;
        float prev_min = 100000000.0;

        // Optimize: Unroll loop to improve performance
        #pragma unroll
        for (int depth = 0; depth < D; depth += D_STEP) {
            prev_min = fminf(dp[ind], prev_min);
            ind += (depth_dim_size * D_STEP);
        }

        MinArray[threadIdx.y][threadIdx.x] = prev_min;
        __syncthreads();

        float d0 = 0, d1 = 0, d2 = 0;

        if (threadIdx.y == 0) {
            int agg_row = threadIdx.x + blockIdx.y * blockDim.y;
            int start_K = K;
            int local_K = 0;

            if (agg_row < m) {
                // Optimize: Use a bounded loop to prevent over-indexing
                for (; (K < (n - 1)) && (K < (start_K + SHMEM_SIZE)); K++) {
                    float d3 = MinArray[threadIdx.x][local_K] + (float) P2;
                    int ind = agg_row * n + K + 1;

                    // Optimize: Use #pragma unroll to potentially unroll inner loop
                    #pragma unroll
                    for (int d = 0; d < D; d += D_STEP) {
                        dp[ind] += cost_image[ind] + dp_criteria(dp, ind - 1, depth_dim_size, d, (float) P1, (float) P2, &d0, &d1, &d2, &d3);
                        ind += (depth_dim_size * D_STEP);
                    }
                    local_K++;
                }
            }
        }

        __syncthreads();
        col += blockDim.x;
    }
}