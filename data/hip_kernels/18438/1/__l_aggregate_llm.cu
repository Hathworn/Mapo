#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//#define DEPTH 2

__device__ float dp_criteria(float *dp, int ind, int depth_dim_size, int d, float P_one, float P_two, float * d_zero, float * d_one, float * d_two, float * d_three) {
    *d_zero = dp[ind];
    if (d > 0)
        *d_one = dp[ind - depth_dim_size] + P_one;
    else
        *d_one = 10000000;

    if (d < D-1)
        *d_two = dp[ind + depth_dim_size] + P_one;
    else
        *d_two = 10000000;
    
    return fminf(fminf(*d_zero, *d_one), fminf(*d_two, *d_three)) - *d_three + P_two;
}

__global__ void __l_aggregate(float *dp, float *cost_image, int m, int n) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = n - 1 - threadIdx.x;
    int depth_dim_size = m * n;
    __shared__ float MinArray[SHMEM_SIZE][SHMEM_SIZE];
    int K = n - 1;

    // Invert the order to ensure correct operation and improve readability, if needed.
    while (col >= 0 && row < m) {
        int ind = row * n + col;
        float prev_min = 100000000.0;
        
        // Optimize loop aggregation for GPU memory access patterns
        for (int depth = 0; depth < D; depth += D_STEP) {
            prev_min = fminf(dp[ind], prev_min);
            ind += (depth_dim_size * D_STEP);
        }

        MinArray[threadIdx.y][SHMEM_SIZE - 1 - threadIdx.x] = prev_min;
        __syncthreads();

        float d0 = 0;
        float d1 = 0;
        float d2 = 0;

        // Only execute for threads with threadIdx.y == 0, optimizing warp execution
        if (threadIdx.y == 0) {
            int agg_row = threadIdx.x + blockIdx.y * blockDim.y;
            int start_K = K;
            int local_K = SHMEM_SIZE - 1;

            if (agg_row < m) {
                for (; K > 0 && K > (start_K - SHMEM_SIZE); K--) {
                    float d3 = MinArray[threadIdx.x][local_K] + (float)P2;
                    int ind = agg_row * n + K - 1;
                    for (int d = 0; d < D; d += D_STEP) {
                        dp[ind] += cost_image[ind] + dp_criteria(dp, ind + 1, depth_dim_size, d, (float)P1, (float)P2, &d0, &d1, &d2, &d3);
                        ind += (depth_dim_size * D_STEP);
                    }
                    local_K--;
                }
            }
        }

        __syncthreads();
        col -= blockDim.x;
    }
}