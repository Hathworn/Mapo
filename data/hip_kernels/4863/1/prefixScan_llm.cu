#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define RAND_RANGE(N) ((double)rand()/((double)RAND_MAX + 1)*(N))

//data generator
__global__ void prefixScan(int* Hist,int* Hist_dev_pre, int noofpartitions,long long size)
{
    extern __shared__ int sharedpartitions[];
    int thd = threadIdx.x;
    int offset = 1;

    // Load input into shared memory
    sharedpartitions[2 * thd] = Hist[2 * thd];
    sharedpartitions[2 * thd + 1] = Hist[2 * thd + 1];

    // Up-sweep (reduce) phase
    for (int i = noofpartitions >> 1; i > 0; i >>= 1) {
        __syncthreads();
        if (thd < i) {
            int x = offset * (2 * thd + 1) - 1;
            int y = offset * (2 * thd + 2) - 1;

            sharedpartitions[y] += sharedpartitions[x];
        }
        offset *= 2;
    }

    // Clear the last element (exclusively)
    if (thd == 0) { 
        sharedpartitions[noofpartitions - 1] = 0; 
    }

    // Down-sweep phase
    for (int i = 1; i < noofpartitions; i *= 2) {
        offset >>= 1;
        __syncthreads();
        if (thd < i) {
            int x = offset * (2 * thd + 1) - 1;
            int y = offset * (2 * thd + 2) - 1;

            int tmp = sharedpartitions[x];
            sharedpartitions[x] = sharedpartitions[y];
            sharedpartitions[y] += tmp;
        }
    }
    __syncthreads();

    // Store the result back to global memory
    Hist_dev_pre[2 * thd] = sharedpartitions[2 * thd];
    Hist_dev_pre[2 * thd + 1] = sharedpartitions[2 * thd + 1];
}