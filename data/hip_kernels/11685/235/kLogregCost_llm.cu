#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Determine correct probability
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 1; // Initialize to 1 to handle the label case efficiently
            for (int i = 0; i < numOut; i++) {
                if (i != label) { // Skip the label position
                    numMax += probs[i * numCases + tx] == maxp;
                }
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}