#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0;

    // Parallel reduction using warp shuffle
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }

    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Reduce the buffer into a single value
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    // Compute gradient using the reduction result
    if (idx == 0) tmp = reduction_buffer[0];
    __syncthreads();

    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}