#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    unsigned int tid = threadIdx.x;
    unsigned int globalIndex = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Loop through elements and accumulate the product
    while (globalIndex < numElements) {
        sum += a[globalIndex] * b[globalIndex];
        globalIndex += gridDim.x * DP_BLOCKSIZE;
    }

    shmem[tid] = sum;
    __syncthreads();

    // Perform reduction in shared memory
    if (DP_BLOCKSIZE >= 512 && tid < 256) shmem[tid] += shmem[tid + 256];
    __syncthreads();
    if (DP_BLOCKSIZE >= 256 && tid < 128) shmem[tid] += shmem[tid + 128];
    __syncthreads();
    if (DP_BLOCKSIZE >= 128 && tid < 64) shmem[tid] += shmem[tid + 64];
    __syncthreads();

    // Reduce last warp using volatile memory operations for performance
    if (tid < 32) {
        volatile float* vsmem = shmem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write the result for this block to the target array
    if (tid == 0) {
        target[blockIdx.x] = shmem[0];
    }
}