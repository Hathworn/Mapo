#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Precompute total number of threads to avoid recomputation
    const int numThreads = blockDim.x * gridDim.x;
    // Loop over elements this thread is responsible for
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Use integer division and modulus to determine the target x, y and corresponding source x, y
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Load and store operation
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}