#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuSearchDoublet( const int* nSpM, const float* spMmat, const int* nSpB, const float* spBmat, const int* nSpT, const float* spTmat, const float* deltaRMin, const float* deltaRMax, const float* cotThetaMax, const float* collisionRegionMin, const float* collisionRegionMax, int* nSpMcomp, int* nSpBcompPerSpM_Max, int* nSpTcompPerSpM_Max, int* nSpBcompPerSpM, int* nSpTcompPerSpM, int* McompIndex, int* BcompIndex, int* tmpBcompIndex, int* TcompIndex, int* tmpTcompIndex) {
    extern __shared__ float sharedMem[];
    int* mPos = (int*)sharedMem;
    int* isMcompat = (int*)&mPos[1];

    if (threadIdx.x == 0) {
        *isMcompat = false;
    }
    __syncthreads();

    float rM = spMmat[blockIdx.x + (*nSpM) * 3];
    float zM = spMmat[blockIdx.x + (*nSpM) * 2];

    int offset = threadIdx.x;
    bool isBcompat;
    bool isTcompat;

    while (offset < max(*nSpB, *nSpT)) {
        isBcompat = threadIdx.x < *nSpB;
        if (isBcompat) {
            float rB = spBmat[offset + (*nSpB) * 3];
            float zB = spBmat[offset + (*nSpB) * 2];
            float deltaR = rM - rB;
            isBcompat = deltaR <= *deltaRMax && deltaR >= *deltaRMin;
            if (isBcompat) {
                float cotTheta = (zM - zB) / deltaR;
                isBcompat = fabsf(cotTheta) <= *cotThetaMax;
                if (isBcompat) {
                    float zOrigin = zM - rM * cotTheta;
                    isBcompat = zOrigin >= *collisionRegionMin && zOrigin <= *collisionRegionMax;
                }
            }
            if (isBcompat) {
                int bPos = atomicAdd(&nSpBcompPerSpM[blockIdx.x], 1);
                tmpBcompIndex[bPos + (*nSpB) * blockIdx.x] = offset;
            }
        }
        
        isTcompat = threadIdx.x < *nSpT;
        if (isTcompat) {
            float rT = spTmat[offset + (*nSpT) * 3];
            float zT = spTmat[offset + (*nSpT) * 2];
            float deltaR = rT - rM;
            isTcompat = deltaR >= *deltaRMin && deltaR <= *deltaRMax;
            if (isTcompat) {
                float cotTheta = (zT - zM) / deltaR;
                isTcompat = fabsf(cotTheta) <= *cotThetaMax;
                if (isTcompat) {
                    float zOrigin = zM - rM * cotTheta;
                    isTcompat = zOrigin >= *collisionRegionMin && zOrigin <= *collisionRegionMax;
                }
            }
            if (isTcompat) {
                int tPos = atomicAdd(&nSpTcompPerSpM[blockIdx.x], 1);
                tmpTcompIndex[tPos + (*nSpT) * blockIdx.x] = offset;
            }
        }

        offset += blockDim.x;
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        if (nSpBcompPerSpM[blockIdx.x] > 0 && nSpTcompPerSpM[blockIdx.x] > 0) {
            *mPos = atomicAdd(nSpMcomp, 1);
            *isMcompat = true;
            McompIndex[*mPos] = blockIdx.x;
            atomicMax(nSpBcompPerSpM_Max, nSpBcompPerSpM[blockIdx.x]);
            atomicMax(nSpTcompPerSpM_Max, nSpTcompPerSpM[blockIdx.x]);
        }
    }

    __syncthreads();

    if (*isMcompat) {
        offset = threadIdx.x;
        while (offset < max(nSpBcompPerSpM[blockIdx.x], nSpTcompPerSpM[blockIdx.x])) {
            if (offset < nSpBcompPerSpM[blockIdx.x]) {
                BcompIndex[offset + (*nSpB) * (*mPos)] = tmpBcompIndex[offset + (*nSpB) * blockIdx.x];
            }
            if (offset < nSpTcompPerSpM[blockIdx.x]) {
                TcompIndex[offset + (*nSpT) * (*mPos)] = tmpTcompIndex[offset + (*nSpT) * blockIdx.x];
            }
            offset += blockDim.x;
        }
    }
}