#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init_rand(hiprandState *states, float* randArr) {
unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
hiprand_init(1234, tid, 0, &states[tid]);  //  Initialize CURAND
randArr[tid] = 2.*M_PI*hiprand_uniform (&states[tid]);     // between 0 and 1
}