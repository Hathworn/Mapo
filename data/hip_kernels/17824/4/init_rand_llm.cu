#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_rand(hiprandState *states, float* randArr) {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;

    // Initialize CURAND state
    hiprandState localState;
    hiprand_init(1234, tid, 0, &localState);

    // Generate random number and store in array
    randArr[tid] = 2.f * M_PI * hiprand_uniform(&localState);
}