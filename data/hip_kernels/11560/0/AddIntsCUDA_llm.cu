#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

// Optimized kernel: Reduce global memory operations and redundant calculations.
__global__ void AddIntsCUDA(int* a, int* b) {
    int local_b = b[0]; // Load b[0] once into a register
    int local_a = a[0]; // Load a[0] once into a register
    
    for (int i = 0; i < 12000000; i++) {
        local_a += local_b; // Use register variables for computation
    }
    
    a[0] = local_a; // Write the result back to global memory once
}