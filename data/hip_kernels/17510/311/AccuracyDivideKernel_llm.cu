#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread 0 to update shared memory for consistent results
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}