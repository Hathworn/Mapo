#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuMeansNoTest(const float * vectsA, size_t na, const float * vectsB, size_t nb, size_t dim, float * means, float * numPairs)
{
    size_t bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x;
    float a, b, sumA = 0.f, sumB = 0.f, localCount = 0.f;

    extern __shared__ float sharedMem[];
    float * threadSumsA = sharedMem;
    float * threadSumsB = sharedMem + NUMTHREADS;
    float * count = sharedMem + 2 * NUMTHREADS;

    if((bx >= na) || (by >= nb))
        return;

    for(size_t offset = tx; offset < dim; offset += NUMTHREADS) {
        a = vectsA[bx * dim + offset];
        b = vectsB[by * dim + offset];

        sumA += a;
        sumB += b;
        localCount += 1.f;
    }

    threadSumsA[tx] = sumA;
    threadSumsB[tx] = sumB;
    count[tx] = localCount;
    __syncthreads();

    for(size_t stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
        if(tx < stride) {
            threadSumsA[tx] += threadSumsA[tx + stride];
            threadSumsB[tx] += threadSumsB[tx + stride];
            count[tx] += count[tx + stride];
        }
        __syncthreads();
    }

    if(tx == 0) {
        means[bx*nb*2+by*2] = threadSumsA[0] / count[0];
        means[bx*nb*2+by*2+1] = threadSumsB[0] / count[0];
        numPairs[bx*nb+by] = count[0];
    }
}