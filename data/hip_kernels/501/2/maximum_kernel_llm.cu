#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void maximum_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // Check bounds for valid element computation
    if(x < n_a && y < n_b) {
        __shared__ float temp[NUM_THREADS];

        // Initialize shared memory
        float local_max = 0.0f;
        for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float t = abs(vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset]);
            local_max = max(local_max, t);
        }
        temp[threadIdx.x] = local_max;

        // Sync with other threads
        __syncthreads();

        // Reduce using shared memory
        for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if(threadIdx.x < stride) {
                temp[threadIdx.x] = max(temp[threadIdx.x], temp[threadIdx.x + stride]);
            }
            __syncthreads();
        }

        // Write the result from the first thread to global memory
        if(threadIdx.x == 0) {
            d[y * pitch_d + x] = temp[0];
        }
    }
}