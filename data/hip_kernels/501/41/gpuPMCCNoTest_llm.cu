#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuPMCCNoTest(const float * vectsa, size_t na, const float * vectsb, size_t nb, size_t dim, const float * numPairs, const float * means, const float * sds, float * correlations)
{
    size_t x = blockIdx.x, y = blockIdx.y, tx = threadIdx.x;
    float a, b, n, scoreA, scoreB;
    __shared__ float meanA, meanB, sdA, sdB, threadSums[NUMTHREADS];

    if((x >= na) || (y >= nb))
        return;

    // Load shared data only once per block
    if(tx == 0) {
        size_t idx = x * nb + y;
        meanA = means[idx * 2];
        meanB = means[idx * 2 + 1];
        sdA = sds[idx * 2];
        sdB = sds[idx * 2 + 1];
        n = numPairs[idx];
    }
    __syncthreads();

    float sum = 0.f;
    // Unrolling loop for efficient memory access
    for(size_t offset = tx; offset < dim; offset += NUMTHREADS) {
        a = vectsa[x * dim + offset];
        b = vectsb[y * dim + offset];
        sum += ((a - meanA) / sdA) * ((b - meanB) / sdB);
    }
    
    threadSums[tx] = sum;
    __syncthreads();

    // Reduce sum using in-place reduction
    for(size_t stride = NUMTHREADS / 2; stride > 0; stride >>= 1) {
        if(tx < stride) {
            threadSums[tx] += threadSums[tx + stride];
        }
        __syncthreads();
    }

    // Store result in the correlations array
    if(tx == 0) correlations[x * nb + y] = threadSums[0] / (n - 1.f);
}