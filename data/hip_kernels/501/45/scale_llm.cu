#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale(float knot_max, int nx, int nsamples, float * x, int pitch_x)
{
    int col_idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Ensure thread does not process columns beyond nx
    if (col_idx >= nx) return;
    
    float min, max, * col = x + col_idx * pitch_x;

    // Optimizing min/max search using a single loop
    min = max = col[0];
    #pragma unroll  // Improve loop execution by unrolling
    for (int i = 1; i < nsamples; i++) {
        float val = col[i];
        if (val < min) min = val;
        if (val > max) max = val;
    }
    
    float delta = max - min;
    
    // Efficient scaling operation
    #pragma unroll  // Better loop throughput via unrolling
    for (int i = 0; i < nsamples; i++) {
        col[i] = (knot_max * (col[i] - min)) / delta;
    }
}