#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuKendall(const float * a, size_t na, const float * b, size_t nb, size_t sampleSize, double * results)
{
    size_t tx = threadIdx.x, ty = threadIdx.y, bx = blockIdx.x, by = blockIdx.y;
    size_t rowa = bx * sampleSize, rowb = by * sampleSize;
    float concordant = 0.f, numer, denom;

    __shared__ float threadSums[NUMTHREADS][NUMTHREADS];

    // Optimized loop
    for (size_t i = tx; i < sampleSize; i += NUMTHREADS) {
        for (size_t j = i + 1 + ty; j < sampleSize; j += NUMTHREADS) {
            int tests = ((a[rowa + j] > a[rowa + i]) && (b[rowb + j] > b[rowb + i]))
                        + ((a[rowa + j] < a[rowa + i]) && (b[rowb + j] < b[rowb + i]))
                        + ((a[rowa + j] == a[rowa + i]) && (b[rowb + j] == b[rowb + i]));
            concordant += (float)tests;
        }
    }

    threadSums[tx][ty] = concordant;
    __syncthreads();

    // Reduce within threads using loop unrolling
    for (size_t i = NUMTHREADS / 2; i > 0; i >>= 1) {
        if (ty < i) {
            threadSums[tx][ty] += threadSums[tx][ty + i];
        }
        __syncthreads();
    }
    
    if (ty == 0) {
        // Reduce across thread blocks
        for (size_t i = NUMTHREADS / 2; i > 0; i >>= 1) {
            if (tx < i) {
                threadSums[tx][0] += threadSums[tx + i][0];
            }
            __syncthreads();
        }
    }

    if ((tx == 0) && (ty == 0)) {
        concordant = threadSums[0][0];
        denom = (float)sampleSize;
        denom = (denom * (denom - 1.f)) / 2.f; 
        float discordant = denom - concordant;
        numer = concordant - discordant;
        results[by * na + bx] = ((double)numer) / ((double)denom);
    }
}