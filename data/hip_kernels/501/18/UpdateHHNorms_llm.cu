#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512

// Updates the column norms by subtracting the Hadamard-square of the
// Householder vector.
//
// N.B.: Overflow incurred in computing the square should already have
// been detected in the original norm construction.

__global__ void UpdateHHNorms(int cols, float *dV, float *dNorms) {
// Copyright 2009, Mark Seligman at Rapid Biologics, LLC.  All rights
// reserved.

    int colIndex = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure thread is within bounds before accessing arrays
    if (colIndex < cols) {
        float val = dV[colIndex];
        
        // Use atomic operation to prevent race conditions if needed
        atomicAdd(&dNorms[colIndex], -val * val);
    }
}