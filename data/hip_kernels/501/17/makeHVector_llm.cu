#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512

__global__ void makeHVector(int rows, float * input, float * output)
{
    int idx = threadIdx.x;
    if (idx >= rows) return;
    
    float sum = 0.f;
    __shared__ float beta;
    __shared__ float sums[NTHREADS];
    
    // Use loop unrolling for better performance
    for (int i = idx; i < rows; i += NTHREADS) {
        if ((idx == 0) && (i == 0)) continue;
        float elt = input[i];
        output[i] = elt;
        sum += elt * elt;
    }
    sums[idx] = sum;
    __syncthreads();

    // Use a more efficient reduction strategy
    for (int i = blockDim.x >> 1; i > 0; i >>= 1) {
        if (idx < i) sums[idx] += sums[idx + i];
        __syncthreads();
    }

    if (idx == 0) {
        float elt = input[0];
        float norm = sqrtf(elt * elt + sums[0]);
        
        elt = (elt > 0) ? elt + norm : elt - norm;
        
        output[0] = elt;
        
        norm = elt * elt + sums[0];
        beta = sqrtf(2.f / norm);
    }
    __syncthreads();

    // Multiply output by beta
    for (int i = idx; i < rows; i += NTHREADS) {
        output[i] *= beta;
    }
}