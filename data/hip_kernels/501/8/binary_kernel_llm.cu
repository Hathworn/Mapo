#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void binary_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // Ensure element indices are within bounds
    if(x < n_a && y < n_b) {
        __shared__ float temp[2 * NUM_THREADS];

        // Initialize shared memory
        temp[threadIdx.x] = 0.0;
        temp[threadIdx.x + NUM_THREADS] = 0.0;

        // Loop with stride to handle more data points
        for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            int a = vg_a[x * pitch_a + offset] != 0.0;
            int b = vg_b[y * pitch_b + offset] != 0.0;
            if(a ^ b) {
                temp[threadIdx.x] += 1.0;
            }
            if(a || b) {
                temp[threadIdx.x + NUM_THREADS] += 1.0;
            }
        }

        // Sync threads before reduction
        __syncthreads();

        // Reduce using a single loop and no shared memory conflicts
        for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if(threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
                temp[threadIdx.x + NUM_THREADS] += temp[threadIdx.x + stride + NUM_THREADS];
            }
            __syncthreads();
        }

        // Write reduced value to global memory
        if(threadIdx.x == 0) {
            float val = temp[0];
            if(temp[NUM_THREADS] != 0.0) {
                val /= temp[NUM_THREADS];
            }
            d[y * pitch_d + x] = val;
        }
    }
}