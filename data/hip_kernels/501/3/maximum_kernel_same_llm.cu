#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void maximum_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x, y = blockIdx.y;

    if (x == y && x < n_a && threadIdx.x == 0) {
        d[y * pitch_d + x] = 0.0;
    }

    if (y < n_a && x < y) {
        __shared__ float temp[NUM_THREADS];
        float max_val = 0.0; // Local variable for max value

        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float t = abs(vg_a[x * pitch_a + offset] - vg_a[y * pitch_a + offset]);
            max_val = max(t, max_val); // Use local variable
        }

        temp[threadIdx.x] = max_val;
        __syncthreads();

        for (size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                temp[threadIdx.x] = max(temp[threadIdx.x], temp[threadIdx.x + stride]);
            }
            __syncthreads();
        }

        if (threadIdx.x == 0) {
            float s = temp[0];
            d[y * pitch_d + x] = s;
            d[x * pitch_d + y] = s;
        }
    }
}