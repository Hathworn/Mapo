#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void canberra_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // Check boundary conditions to avoid unnecessary computations
    if (x < n_a && y < n_b) {
        __shared__ float temp[NUM_THREADS];

        // Initialize shared memory
        temp[threadIdx.x] = 0.0f;
        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float num = fabsf(vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset]);
            float den = fabsf(vg_a[x * pitch_a + offset] + vg_b[y * pitch_b + offset]);
            if (den != 0.0f) {
                temp[threadIdx.x] += num / den;
            }
        }

        // Sync threads to ensure completion of all computations 
        __syncthreads();

        // Perform parallel reduction
        for (size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            // Synchronize to make sure all threads have updated their temp values
            __syncthreads();
        }

        // Write the result of reduction to global memory
        if (threadIdx.x == 0) {
            d[y * pitch_d + x] = temp[0];
        }
    }
}