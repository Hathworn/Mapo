#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuSDNoTest(const float * vectsA, size_t na, const float * vectsB, size_t nb, size_t dim, const float * means, const float * numPairs, float * sds)
{
    size_t tx = threadIdx.x;
    size_t bx = blockIdx.x, by = blockIdx.y;
    float threadSumA = 0.f, threadSumB = 0.f;
    __shared__ float meanA, meanB, n;

    if((bx >= na) || (by >= nb))
        return;

    if(tx == 0) {
        meanA = means[bx * nb * 2 + by * 2];
        meanB = means[bx * nb * 2 + by * 2 + 1];
        n = numPairs[bx * nb + by];
    }
    __syncthreads();

    for(size_t offset = tx; offset < dim; offset += NUMTHREADS) {
        float a = vectsA[bx * dim + offset];
        float b = vectsB[by * dim + offset];
        float termA = a - meanA;
        float termB = b - meanB;
        threadSumA += termA * termA;
        threadSumB += termB * termB;
    }
    __shared__ float threadSumsA[NUMTHREADS], threadSumsB[NUMTHREADS];
    threadSumsA[tx] = threadSumA;
    threadSumsB[tx] = threadSumB;
    __syncthreads();

    for(size_t stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
        if(tx < stride) {
            threadSumsA[tx] += threadSumsA[tx + stride];
            threadSumsB[tx] += threadSumsB[tx + stride];
        }
        __syncthreads();
    }

    if(tx == 0) {
        sds[bx * nb * 2 + by * 2]     = sqrtf(threadSumsA[0] / (n - 1.f));
        sds[bx * nb * 2 + by * 2 + 1] = sqrtf(threadSumsB[0] / (n - 1.f));
    }
}