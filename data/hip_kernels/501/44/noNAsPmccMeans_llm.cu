#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void noNAsPmccMeans(int nRows, int nCols, float * a, float * means) {
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    if (col >= nCols) return;
    
    int inOffset = col * nRows;
    float sum = 0.f;

    __shared__ float threadSums[NUMTHREADS];

    // Unroll the loop for efficiency
    for (int i = threadIdx.y; i < nRows; i += blockDim.y) {
        sum += a[inOffset + i];
    }
    
    // Reduce sum within the block with improved stride access
    int tid = threadIdx.y;
    threadSums[tid] = sum;
    __syncthreads();

    for (int i = blockDim.y >> 1; i > 0; i >>= 1) {
        if (tid < i) {
            threadSums[tid] += threadSums[tid + i];
        }
        __syncthreads();
    }

    if (tid == 0) {
        means[col] = threadSums[0] / (float)nRows;
    }
}