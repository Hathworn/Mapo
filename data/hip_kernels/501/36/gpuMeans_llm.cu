#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuMeans(const float *vectsA, size_t na, const float *vectsB, size_t nb, size_t dim, float *means, float *numPairs) {
    size_t bx = blockIdx.x, by = blockIdx.y, tx = threadIdx.x;
    float a, b;

    // Reduced shared memory usage by using one array with structures
    __shared__ struct {
        float sumA;
        float sumB;
        float count;
    } smem[NUMTHREADS];

    if (bx >= na || by >= nb) return;

    smem[tx].sumA = smem[tx].sumB = smem[tx].count = 0.f;

    for (size_t offset = tx; offset < dim; offset += NUMTHREADS) {
        a = vectsA[bx * dim + offset];
        b = vectsB[by * dim + offset];
        if (!isnan(a) && !isnan(b)) {
            smem[tx].sumA += a;
            smem[tx].sumB += b;
            smem[tx].count += 1.f;
        }
    }
    __syncthreads();

    for (size_t stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
        if (tx < stride) {
            smem[tx].sumA += smem[tx + stride].sumA;
            smem[tx].sumB += smem[tx + stride].sumB;
            smem[tx].count += smem[tx + stride].count;
        }
        __syncthreads();
    }

    if (tx == 0) {
        means[bx * nb * 2 + by * 2] = smem[0].sumA / smem[0].count;
        means[bx * nb * 2 + by * 2 + 1] = smem[0].sumB / smem[0].count;
        numPairs[bx * nb + by] = smem[0].count;
    }
}