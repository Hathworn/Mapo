#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512

__global__ void getColNorms(int rows, int cols, float* da, int lda, float* colNorms)
{
    int colIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (colIndex >= cols) return;

    float sum = 0.f;  // Initialize the sum

    // Use shared memory to reduce global memory accesses
    __shared__ float partialSum[NTHREADS];  
    partialSum[threadIdx.x] = 0.f;  // Initialize shared memory

    float* col = da + colIndex * lda;
    for (int i = threadIdx.x; i < rows; i += blockDim.x) {
        float term = col[i];
        partialSum[threadIdx.x] += term * term;
    }

    // Use a parallel reduction to sum up squared terms
    __syncthreads();
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            partialSum[threadIdx.x] += partialSum[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // The result is stored in the first thread of each block
    if (threadIdx.x == 0) {
        colNorms[colIndex] = partialSum[0];
    }
}