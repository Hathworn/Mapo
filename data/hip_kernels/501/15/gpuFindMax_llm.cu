#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512

// Updates the column norms by subtracting the Hadamard-square of the
// Householder vector.
//
// N.B.:  Overflow incurred in computing the square should already have
// been detected in the original norm construction.

__global__ void gpuFindMax(int n, float *data, int threadWorkLoad, int *maxIndex)
{
    int start = threadWorkLoad * threadIdx.x;
    int end = start + threadWorkLoad;

    // Shared memory for maximum indices per thread
    __shared__ int maxIndices[NTHREADS];
    maxIndices[threadIdx.x] = -1;

    // Check if start is out of bounds
    if (start >= n) return;

    // Initialize local max index
    int localMaxIndex = start;
    
    // Find local maximum index within thread's workload
    for (int i = start + 1; i < end && i < n; i++) {
        if (data[i] > data[localMaxIndex]) {
            localMaxIndex = i;
        }
    }
    maxIndices[threadIdx.x] = localMaxIndex;
    __syncthreads();

    // Reduce to find the maximum index in the block
    for (int offset = blockDim.x >> 1; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            int j = maxIndices[threadIdx.x];
            int k = maxIndices[threadIdx.x + offset];
            if (data[j] < data[k]) {
                maxIndices[threadIdx.x] = k;
            }
        }
        __syncthreads();
    }

    // Write the result for the block
    if (threadIdx.x == 0) {
        *maxIndex = maxIndices[0];
    }
}