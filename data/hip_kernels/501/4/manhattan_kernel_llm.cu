#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void manhattan_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // If all elements need to be computed
    if(x < n_a && y < n_b) {
        __shared__ float temp[NUM_THREADS];

        // Initialize shared memory and calculate partial sums
        float sum = 0.0;
        for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float t = fabsf(vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset]); // Use fabsf for float
            sum += t; // Accumulate in register
        }
        temp[threadIdx.x] = sum;
        __syncthreads(); // Ensure all writes to shared memory are done

        // Reduction in shared memory
        for(int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if(threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Write result to global memory
        if(threadIdx.x == 0) {
            d[y * pitch_d + x] = temp[0];
        }
    }
}