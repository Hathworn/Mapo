#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
#define max(a, b) ((a > b)?a:b)

#define THREADSPERDIM   16

#define FALSE 0
#define TRUE !FALSE

__global__ void getUnrestricted(int countx, int county, int rows, int cols, float * mX, int mXdim, float * vY, int vYdim, float * mQ, int mQdim, float * mR, int mRdim, float * vectB, int vectBdim) {

    int n = blockIdx.x * THREADSPERDIM + threadIdx.x;
    int m = blockIdx.y * THREADSPERDIM + threadIdx.y;
    int i, j, k;
    float sum, invnorm;
    float *X, *Y, *Q, *R, *B;
    float *coli, *colj, *colQ, *colX;
    
    if((m >= county) || (n >= countx)) return;

    X = mX + (m * countx + n) * mXdim;
    Y = vY + (m*countx+n) * vYdim;
    B = vectB + (m*countx+n) * vectBdim;
    Q = mQ + (m*countx+n) * mQdim;
    R = mR + (m*countx+n) * mRdim;

    // Unroll initialization of intercepts
    for(i = 0; i < rows; i += 4) {
        X[i] = 1.f;
        if (i + 1 < rows) X[i+1] = 1.f;
        if (i + 2 < rows) X[i+2] = 1.f;
        if (i + 3 < rows) X[i+3] = 1.f;
    }

    // Initialize Q with X ...
    for(i = 0; i < rows; i++) {
        for(j = 0; j < cols; j++)
            Q[i+j*rows] = X[i+j*rows];
    }

    // Gram-Schmidt Process to find Q
    for(j = 0; j < cols; j++) {
        colj = Q+rows*j;
        for(i = 0; i < j; i++) {
            coli = Q+rows*i;
            sum = 0.f;
            for(k = 0; k < rows; k++)
                sum += coli[k] * colj[k];

            // Use sum in a single loop for modification
            for(k = 0; k < rows; k++)
                colj[k] -= sum * coli[k];
        }
        
        sum = 0.f;
        for(i = 0; i < rows; i++)
            sum += colj[i] * colj[i];
        invnorm = 1.f / sqrtf(sum);

        // Vectorize multiplication by invnorm
        for(i = 0; i < rows; i++)
            colj[i] *= invnorm;
    }

    for(i = cols-1; i >= 0; i--) {
        colQ = Q+i*rows;
        
        // Matrix multiplication Q * X -> R
        for(j = 0; j < cols; j++) {
            colX = X+j*rows;
            sum = 0.f;
            for(k = 0; k < rows; k++)
                sum += colQ[k] * colX[k];
            R[i+j*cols] = sum;
        }
        
        sum = 0.f;
        // Compute the vector Q^t * Y -> B
        for(j = 0; j < rows; j++)
            sum += colQ[j] * Y[j];
        
        // Back substitution to find the x for Rx = B
        for(j = cols-1; j > i; j--)
            sum -= R[i+j*cols] * B[j];

        B[i] = sum / R[i+i*cols];
    }
}