#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuSignif(const float * gpuNumPairs, const float * gpuCorrelations, size_t n, float * gpuTScores)
{
    // Calculate global thread ID
    const size_t global_tid = blockIdx.x * blockDim.x * THREADWORK + threadIdx.x * THREADWORK;
    
    for (int i = 0; i < THREADWORK; ++i) {
        size_t idx = global_tid + i;
        if (idx >= n)
            break;

        float npairs = gpuNumPairs[idx];
        float cor = gpuCorrelations[idx];
        // Precompute common expression
        float radicand = (npairs - 2.f) / (1.f - cor * cor);
        // Directly write to global memory
        gpuTScores[idx] = cor * sqrtf(radicand);
    }
}