#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void binary_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // Initialize diagonal elements
    if(x == y && x < n_a && threadIdx.x == 0) {
        d[y * pitch_d + x] = 0.0;
    }

    // Calculate elements only if y < n_a and x < y
    if(y < n_a && x < y) {
        __shared__ float temp[2 * NUM_THREADS];

        float local_diff = 0.0;
        float local_union = 0.0;

        // Use loop unrolling for efficiency
        for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x * 2) {
            int a = vg_a[x * pitch_a + offset] != 0.0;
            int b = vg_a[y * pitch_a + offset] != 0.0;
            if(a ^ b) local_diff += 1.0;
            if(a || b) local_union += 1.0;

            if(offset + blockDim.x < k) {
                int a_next = vg_a[x * pitch_a + offset + blockDim.x] != 0.0;
                int b_next = vg_a[y * pitch_a + offset + blockDim.x] != 0.0;
                if(a_next ^ b_next) local_diff += 1.0;
                if(a_next || b_next) local_union += 1.0;
            }
        }
        
        temp[threadIdx.x] = local_diff;
        temp[threadIdx.x + NUM_THREADS] = local_union;

        __syncthreads();

        // Loop unrolling for reduction
        for(size_t stride = blockDim.x / 2; stride > 0; stride >>= 1) {
            if(threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
                temp[threadIdx.x + NUM_THREADS] += temp[threadIdx.x + stride + NUM_THREADS];
            }
            __syncthreads();
        }

        if(threadIdx.x == 0) {
            float val = temp[0];
            if(temp[NUM_THREADS] != 0.0) {
                val /= temp[NUM_THREADS];
            }
            d[y * pitch_d + x] = val;
            d[x * pitch_d + y] = val;
        }
    }
}