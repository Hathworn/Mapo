#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NTHREADS 512

__global__ void gpuSwapCol(int rows, float * dArray, int coli, int * dColj, int * dPivot)
{
    int rowIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds
    if(rowIndex >= rows)
        return;

    int colj = coli + (*dColj);
    
    // Pre-calculate memory offsets
    int offsetColi = rowIndex + coli * rows;
    int offsetColj = rowIndex + colj * rows;

    // Swap elements
    float fholder = dArray[offsetColi];
    dArray[offsetColi] = dArray[offsetColj];
    dArray[offsetColj] = fholder;

    // Swap pivot entries using a single thread
    if((blockIdx.x == 0) && (threadIdx.x == 0)) {
        int iholder = dPivot[coli];
        dPivot[coli] = dPivot[colj];
        dPivot[colj] = iholder;
    }
}