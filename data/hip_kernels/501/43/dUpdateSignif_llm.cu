#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__device__ int dIsSignificant(float signif, int df) {
    float tcutoffs[49] = {
        637.000, 31.600, 2.920, 8.610, 6.869, 5.959, 5.408, 5.041, 4.781,
        4.587, 4.437, 4.318, 4.221, 4.140, 4.073, 4.015, 3.965, 3.922,
        3.883, 3.850, 3.819, 3.792, 3.768, 3.745, 3.725, 3.707, 3.690,
        3.674, 3.659, 3.646,
        3.622, 3.601, 3.582, 3.566, 3.551, 3.538, 3.526, 3.515, 3.505, 3.496,
        3.476, 3.460, 3.447, 3.435,
        3.416, // 80
        3.390, // 100
        3.357, // 150
        3.340, // 200
        3.290  // > 200
    };

    size_t index;
    if (df <= 0) return 0;
    if (df <= 30) index = df - 1;
    else if (df <= 50) index = 30 + (df + (df % 2) - 32) / 2;
    else if (df <= 70) index = 40 + (df - 55) / 5;
    else if (df <= 80) index = 44;
    else if (df <= 100) index = 45;
    else if (df <= 150) index = 46;
    else if (df <= 200) index = 47;
    else index = 48;

    return fabsf(signif) < tcutoffs[index] ? FALSE : TRUE;
}

__global__ void dUpdateSignif(const float *gpuData, size_t n, float *gpuResults) {
    size_t bx = blockIdx.x, tx = threadIdx.x;
    size_t start = bx * NUMTHREADS * THREADWORK + tx * THREADWORK;

    for (size_t i = 0; i < THREADWORK; i++) {
        size_t idx = start + i;
        if (idx >= n) break; // Avoid out-of-bound access

        size_t inrow = idx * 5;
        size_t outrow = idx * 6;

        float cor = gpuData[inrow + 3];
        float npairs = gpuData[inrow + 4];
        float tscore;

        if (cor >= 0.999f) 
            tscore = 10000.0f;
        else {
            float radicand = (npairs - 2.f) / (1.f - cor * cor);
            tscore = cor * sqrtf(radicand);
        }

        if (dIsSignificant(tscore, (int)npairs)) {
            gpuResults[outrow] = gpuData[inrow];
            gpuResults[outrow + 1] = gpuData[inrow + 1];
            gpuResults[outrow + 2] = gpuData[inrow + 2];
            gpuResults[outrow + 3] = cor;
            gpuResults[outrow + 4] = tscore;
            gpuResults[outrow + 5] = npairs;
        } else {
            gpuResults[outrow] = -1.f;
        }
    }
}