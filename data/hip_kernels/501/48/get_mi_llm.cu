#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void get_mi(int nbins, int nsamples, int nx, float * x_bin_scores, int pitch_x_bin_scores, float * entropies_x, int ny, float * y_bin_scores, int pitch_y_bin_scores, float * entropies_y, float * mis, int pitch_mis)
{
    int col_x = blockDim.x * blockIdx.x + threadIdx.x;
    int col_y = blockDim.y * blockIdx.y + threadIdx.y;

    if (col_x >= nx || col_y >= ny) return;

    float mi = 0.f;
    float *x_bins = x_bin_scores + col_x * pitch_x_bin_scores;
    float *y_bins = y_bin_scores + col_y * pitch_y_bin_scores;

    // Use shared memory to improve memory access efficiency
    extern __shared__ float shared_mem[];
    float *shared_x_bins = shared_mem;
    float *shared_y_bins = shared_mem + nbins * nsamples;

    for (int i = 0; i < nbins; i++) {
        for (int k = 0; k < nsamples; k++) {
            shared_x_bins[k * nbins + i] = x_bins[k * nbins + i];
            shared_y_bins[k * nbins + i] = y_bins[k * nbins + i];
        }
    }
    __syncthreads();

    // Calculate joint entropy
    for (int i = 0; i < nbins; i++) {
        for (int j = 0; j < nbins; j++) {
            float prob = 0.f;
            for (int k = 0; k < nsamples; k++) {
                prob += shared_x_bins[k * nbins + i] * shared_y_bins[k * nbins + j];
            }
            prob /= (float)nsamples;

            float logp = (prob <= 0.f) ? 0.f : __log2f(prob);
            mi += prob * logp;
        }
    }

    // Calculate mi from entropies
    mi += entropies_x[col_x] + entropies_y[col_y];
    (mis + col_y * pitch_mis)[col_x] = mi;
}