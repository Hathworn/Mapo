#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
#define max(a, b) ((a > b)?a:b)

#define THREADSPERDIM   16

#define FALSE 0
#define TRUE !FALSE

__global__ void ftest(int diagFlag, int p, int rows, int colsx, int colsy, int rCols, int unrCols, float * obs, int obsDim, float * rCoeffs, int rCoeffsDim, float * unrCoeffs, int unrCoeffsDim, float * rdata, int rdataDim, float * unrdata, int unrdataDim, float * dfStats) 
{
    int j = blockIdx.x * THREADSPERDIM + threadIdx.x;
    int i = blockIdx.y * THREADSPERDIM + threadIdx.y;
    int idx = i * colsx + j;

    if ((i >= colsy) || (j >= colsx)) return;
    if ((!diagFlag) && (i == j)) {
        dfStats[idx] = 0.f;
        return;
    }

    float kobs, fp = (float)p, frows = (float)rows, rSsq = 0.f, unrSsq = 0.f, rEst, unrEst, score = 0.f;
    
    // Pre-calculate row indices
    float* tObs = obs + (i * colsx + j) * obsDim;
    float* tRCoeffs = rCoeffs + i * rCoeffsDim;
    float* tRdata = rdata + i * rdataDim;
    float* tUnrCoeffs = unrCoeffs + (i * colsx + j) * unrCoeffsDim;
    float* tUnrdata = unrdata + (i * colsx + j) * unrdataDim;

    // Optimize loop by minimizing calculations inside loops
    for (int k = 0; k < rows; ++k) {
        rEst = 0.f;
        unrEst = 0.f;
        kobs = tObs[k];

        // Unroll inner loops
        for (int m = 0; m < rCols; ++m) {
            rEst += tRCoeffs[m] * tRdata[k + m * rows];
        }
        for (int m = 0; m < unrCols; ++m) {
            unrEst += tUnrCoeffs[m] * tUnrdata[k + m * rows];
        }

        rSsq += (kobs - rEst) * (kobs - rEst);
        unrSsq += (kobs - unrEst) * (kobs - unrEst);
    }

    score = ((rSsq - unrSsq) * (frows - 2.f * fp - 1.f)) / (fp * unrSsq);

    if (!isfinite(score)) {
        score = 0.f;
    }

    dfStats[idx] = score;
}