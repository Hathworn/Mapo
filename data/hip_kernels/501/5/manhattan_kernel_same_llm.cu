#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void manhattan_kernel_same(
    const float *vg_a, size_t pitch_a, size_t n_a,
    const float *vg_b, size_t pitch_b, size_t n_b,
    size_t k, float *d, size_t pitch_d, float p) {

    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // Initialize distance to zero when x equals y
    if (x == y && x < n_a && threadIdx.x == 0) {
        d[y * pitch_d + x] = 0.0;
    }

    // Ensure within bounds before computation
    if (y < n_a && x < y) {
        __shared__ float temp[NUM_THREADS];
        float local_sum = 0.0;  // Local sum for each thread

        // Calculate partial distances
        for (size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float t = fabsf(vg_a[x * pitch_a + offset] - vg_a[y * pitch_a + offset]);
            local_sum += t;
        }
        temp[threadIdx.x] = local_sum;

        // Sync after initialization
        __syncthreads();

        // Reduction to accumulate results
        for (size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Store the results in global memory
        if (threadIdx.x == 0) {
            float s = temp[0];
            d[y * pitch_d + x] = s;
            d[x * pitch_d + y] = s;
        }
    }
}