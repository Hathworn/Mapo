#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuSD(const float *vectsA, size_t na, const float *vectsB, size_t nb, size_t dim, const float *means, const float *numPairs, float *sds) {
    size_t tx = threadIdx.x;
    size_t bx = blockIdx.x, by = blockIdx.y;
    float a, b, termA, termB;
    __shared__ float meanA, meanB, n;
    __shared__ float threadSumsA[NUMTHREADS], threadSumsB[NUMTHREADS];

    if ((bx >= na) || (by >= nb)) return; // Boundary check

    // Load shared memory for mean and n
    if (tx == 0) {
        meanA = means[bx * nb * 2 + by * 2];
        meanB = means[bx * nb * 2 + by * 2 + 1];
        n = numPairs[bx * nb + by];
    }
    __syncthreads();

    // Initialize shared thread sums
    threadSumsA[tx] = 0.f;
    threadSumsB[tx] = 0.f;
    
    // Calculate partial sums
    for (size_t offset = tx; offset < dim; offset += NUMTHREADS) {
        a = vectsA[bx * dim + offset];
        b = vectsB[by * dim + offset];
        // Ensure valid elements
        if (!(isnan(a) || isnan(b))) {
            termA = a - meanA;
            termB = b - meanB;
            threadSumsA[tx] += termA * termA;
            threadSumsB[tx] += termB * termB;
        }
    }
    __syncthreads();

    // Reduce sums
    for (size_t stride = NUMTHREADS / 2; stride > 0; stride >>= 1) {
        if (tx < stride) {
            threadSumsA[tx] += threadSumsA[tx + stride];
            threadSumsB[tx] += threadSumsB[tx + stride];
        }
        __syncthreads();
    }
    
    // Write result
    if (tx == 0) {
        sds[bx * nb * 2 + by * 2]     = sqrtf(threadSumsA[0] / (n - 1.f));
        sds[bx * nb * 2 + by * 2 + 1] = sqrtf(threadSumsB[0] / (n - 1.f));
    }
}