#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define max(a, b) ((a > b)?a:b)

#define THREADSPERDIM   16

#define FALSE 0
#define TRUE !FALSE

// mX has order rows x cols
// vectY has length rows

__global__ void getRestricted(int countx, int county, int rows, int cols, float * mX, int mXdim, float * vY, int vYdim, float * mQ, int mQdim, float * mR, int mRdim, float * vectB, int vectBdim) {

    int m = blockIdx.x * THREADSPERDIM + threadIdx.x;
    if (m >= county) return;

    int n = (m == 1) ? 0 : 1;

    float *X = mX + (m * mXdim);
    float *Y = vY + (m * countx + n) * vYdim;
    float *B = vectB + m * vectBdim;
    float *Q = mQ + m * mQdim;
    float *R = mR + m * mRdim;

    // Initialize intercepts
    for (int i = 0; i < rows; i++) 
        X[i] = 1.f;

    // Initialize Q with X
    for (int i = 0; i < rows; i++) {
#pragma unroll
        for (int j = 0; j < cols; j++)
            Q[i + j * rows] = X[i + j * rows];
    }

    // Gram-Schmidt process to find Q
    for (int j = 0; j < cols; j++) {
        float *colj = Q + rows * j;
        for (int i = 0; i < j; i++) {
            float *coli = Q + rows * i;
            float sum = 0.f;

            // Sum = dot product of coli and colj
            for (int k = 0; k < rows; k++) 
                sum += coli[k] * colj[k];
            
            for (int k = 0; k < rows; k++)
                colj[k] -= sum * coli[k];
        }
        // Normalize colj
        float sum = 0.f;
        for (int i = 0; i < rows; i++)
            sum += colj[i] * colj[i];
        float invnorm = 1.f / sqrtf(sum);
        for (int i = 0; i < rows; i++)
            colj[i] *= invnorm;
    }

    // Matrix multiplication Q * X -> R and QR decomposition
    for (int i = cols - 1; i >= 0; i--) {
        float *colQ = Q + i * rows;
        for (int j = 0; j < cols; j++) {
            float *colX = X + j * rows;
            float sum = 0.f;
            // Sum = dot product of colQ and colX
            for (int k = 0; k < rows; k++)
                sum += colQ[k] * colX[k];
            R[i + j * cols] = sum;
        }
        float sum = 0.f;
        // Compute vector Q^t * Y -> B
        for (int j = 0; j < rows; j++)
            sum += colQ[j] * Y[j];
        // Back substitution for Rx = B
        for (int j = cols - 1; j > i; j--)
            sum -= R[i + j * cols] * B[j];
        
        B[i] = sum / R[i + i * cols];
    }
}