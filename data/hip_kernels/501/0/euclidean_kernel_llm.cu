#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void euclidean_kernel(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;

    // If an element is to be computed
    if(x < n_a && y < n_b) {
        __shared__ float temp[NUM_THREADS];

        float sum = 0.0f; // Use a register for partial sum
        for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float t = vg_a[x * pitch_a + offset] - vg_b[y * pitch_b + offset];
            sum += (t * t);
        }
        temp[threadIdx.x] = sum;

        // Sync with other threads
        __syncthreads();

        // Unroll and optimize reduction
        for(size_t stride = blockDim.x / 2; stride > 0; stride /= 2) {
            if(threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Write to global memory
        if(threadIdx.x == 0) {
            d[y * pitch_d + x] = sqrt(temp[0]);
        }
    }
}