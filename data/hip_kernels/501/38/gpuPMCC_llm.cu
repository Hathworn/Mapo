#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE !FALSE

#define NUMTHREADS 16
#define THREADWORK 32

__global__ void gpuPMCC(const float * vectsa, size_t na, const float * vectsb, size_t nb, size_t dim, const float * numPairs, const float * means, const float * sds, float * correlations)
{
    size_t x = blockIdx.x, y = blockIdx.y, tx = threadIdx.x;
    float a, b, n, scoreA, scoreB;
    __shared__ float meanA, meanB, sdA, sdB;
    __shared__ float threadSums[NUMTHREADS];

    if((x >= na) || (y >= nb))
        return;

    // Initialize shared values for better memory performance
    if(tx == 0) {
        meanA = means[x*nb*2+y*2];
        meanB = means[x*nb*2+y*2+1];
        sdA = sds[x*nb*2+y*2];
        sdB = sds[x*nb*2+y*2+1];
        n = numPairs[x*nb+y];
    }
    __syncthreads();

    // Initialize sum for this thread
    float sum = 0.f;
    for(size_t offset = tx; offset < dim; offset += NUMTHREADS) {
        a = vectsa[x * dim + offset];
        b = vectsb[y * dim + offset];
        if(!(isnan(a) || isnan(b))) {
            scoreA = (a - meanA) / sdA;
            scoreB = (b - meanB) / sdB;
            sum += scoreA * scoreB;
        }
    }
    threadSums[tx] = sum;
    __syncthreads();

    // Reduction to accumulate results
    for(size_t stride = NUMTHREADS >> 1; stride > 0; stride >>= 1) {
        if(tx < stride) 
            threadSums[tx] += threadSums[tx + stride];
        __syncthreads();
    }
    if(tx == 0) 
        correlations[x*nb+y] = threadSums[0] / (n - 1.f);
}