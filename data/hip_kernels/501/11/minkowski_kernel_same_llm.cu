#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define NUM_THREADS 32

__global__ void minkowski_kernel_same(const float * vg_a, size_t pitch_a, size_t n_a, const float * vg_b, size_t pitch_b, size_t n_b, size_t k, float * d, size_t pitch_d, float p)
{
    size_t x = blockIdx.x;
    size_t y = blockIdx.y;
    
    // Initialize shared memory for parallel reduction
    __shared__ float temp[NUM_THREADS];
    
    if(x == y && x < n_a && threadIdx.x == 0) {
        d[y * pitch_d + x] = 0.0;
    }

    if(y < n_a && x < y) {
        float sum = 0.0;
        
        // Each thread accumulates part of the sum
        for(size_t offset = threadIdx.x; offset < k; offset += blockDim.x) {
            float t = fabsf(vg_a[x * pitch_a + offset] - vg_a[y * pitch_a + offset]);
            sum += __powf(t, p);
        }
        
        temp[threadIdx.x] = sum; // Store partial sums in shared memory
        
        __syncthreads(); // Ensure all threads have written their partial sums
        
        // Parallel reduction to sum partial results
        for(size_t stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
            if(threadIdx.x < stride) {
                temp[threadIdx.x] += temp[threadIdx.x + stride];
            }
            __syncthreads(); // Ensure all threads are synchronized
        }
        
        // Write the result to global memory
        if(threadIdx.x == 0) {
            float power = 1.f / p;
            float s = __powf(temp[0], power);
            d[y * pitch_d + x] = s;
            d[x * pitch_d + y] = s;
        }
    }
}