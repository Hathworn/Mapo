#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float do_fraction(float numer, float denom) {
    float result = 0.f;
    if((numer == denom) && (numer != 0.f))
        result = 1.f;
    else if(denom != 0.f)
        result = numer / denom;
    return result;
}

__global__ void get_bin_scores(int nbins, int order, int nknots, float *knots, int nsamples, int nx, float *x, int pitch_x, float *bins, int pitch_bins) {
    int col_x = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if the thread is within the bounds
    if(col_x >= nx) return;

    float ld, rd, z, term1, term2;
    float *in_col = x + col_x * pitch_x;
    float *bin_col = bins + col_x * pitch_bins;

    for(int k = 0; k < nsamples; k++, bin_col += nbins) {
        z = in_col[k];
        int i0 = min(nbins - 1, max(0, (int)floorf(z) + order - 1)); // Ensure i0 is within bounds

        bin_col[i0] = 1.f;
        for(int i = 2; i <= order; i++) {
            // Iterate over range ensuring j is always in valid range
            for(int j = max(0, i0 - i + 1); j <= min(i0, nbins - 1); j++) {
                if(j < nknots && j + i - 1 < nknots) {
                    ld = do_fraction(z - knots[j], knots[j + i - 1] - knots[j]);
                    term1 = ld * bin_col[j];
                } else {
                    term1 = 0.f;
                }
                
                if(j + 1 < nknots && j + i < nknots) {
                    rd = do_fraction(knots[j + i] - z, knots[j + i] - knots[j + 1]);
                    term2 = rd * bin_col[j + 1];
                } else {
                    term2 = 0.f;
                }
                
                // Combine terms to update bin_col
                bin_col[j] = term1 + term2;
            }
        }
    }
}