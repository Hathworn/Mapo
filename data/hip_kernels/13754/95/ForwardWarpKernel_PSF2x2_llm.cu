#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Pre-compute the coordinates of the corner and the pixel shift
    float cx = u[flow_row_offset + j] * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + static_cast<float>(i) + 1.0f;

    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);
    float value = src[image_row_offset + j];

    // Use shared bound check logic
    auto withinBounds = [w, h](int x, int y) { return !(x >= w || x < 0 || y >= h || y < 0); };

    struct {
      float wx, wy;
      int ty_adjust, tx_adjust;
    } corners[4] = {
      {dx, dy, 0, 0},            // bottom-right
      {1.0f - dx, dy, 0, -1},    // bottom-left
      {1.0f - dx, 1.0f - dy, -1, -1}, // upper-left
      {dx, 1.0f - dy, -1, 0}     // upper-right
    };

    for (const auto& corner : corners) {
        int current_tx = tx + corner.tx_adjust;
        int current_ty = ty + corner.ty_adjust;
        if (withinBounds(current_tx, current_ty)) {
            float weight = corner.wx * corner.wy;
            _atomicAdd(dst + current_ty * image_stride + current_tx, value * weight);
            _atomicAdd(normalization_factor + current_ty * image_stride + current_tx, weight);
        }
    }
}