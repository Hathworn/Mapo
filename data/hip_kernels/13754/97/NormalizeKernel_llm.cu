#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit for threads out of bounds
    if (i >= h || j >= w) return;

    const int pos = i * s + j;

    float scale = normalization_factor[pos];

    // Use fmaf for potential higher precision and to avoid branching
    image[pos] = __fmul_rn(image[pos], (scale == 0.0f ? 1.0f : __frcp_rn(scale)));
}