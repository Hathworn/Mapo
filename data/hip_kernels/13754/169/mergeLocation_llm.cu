#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate element index with grid size to improve scalability
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop through all points with stride to use more threads efficiently
    for (int ptidx = idx; ptidx < npoints; ptidx += stride)
    {
        short2 loc = loc_[ptidx];
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}