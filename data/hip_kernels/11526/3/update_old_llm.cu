#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_old(float4 *__restrict__ newPos, float4 *__restrict__ oldPos) {
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to coalesce global memory access
    __shared__ float4 sharedPos[256]; // Adjust size according to blockDim.x if necessary

    // Load data from global to shared memory
    sharedPos[threadIdx.x] = newPos[index];

    __syncthreads(); // Ensure all threads have loaded data

    // Write data from shared to global memory
    oldPos[index] = sharedPos[threadIdx.x];
}