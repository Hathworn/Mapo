#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vel_step( float4 *__restrict__ deviceVel, float3 *__restrict__ accels, unsigned int numBodies, float dt)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Check if the index is within bounds
    if (index >= numBodies) { return; }
    
    // Update the velocity by combining operations
    float half_dt = 0.5f * dt;
    float3 accel = accels[index];
    deviceVel[index].x += accel.x * half_dt;
    deviceVel[index].y += accel.y * half_dt;
    deviceVel[index].z += accel.z * half_dt;
}