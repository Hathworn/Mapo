#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void r_step( float4 *__restrict__ devPos, float4 *__restrict__ deviceVel, unsigned int numBodies, float dt)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is less than numBodies to prevent out-of-bounds access
    if (index >= numBodies) { return; }
    // Use temporary variable to reduce multiple memory accesses
    float4 velocity = deviceVel[index];
    devPos[index].x += velocity.x * dt;
    devPos[index].y += velocity.y * dt;
    devPos[index].z += velocity.z * dt;
}