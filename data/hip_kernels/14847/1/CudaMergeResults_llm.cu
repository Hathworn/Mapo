#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//#define DEBUG
//#define HANDLE_ERROR(x) if((x) != 0) cout << "Error!" << endl;

using namespace std;

struct SubBlock{
    int * nnz_global_i_idx;
    int * nnz_global_o_idx;
    int nnz;
    int * nnz_local_r_idx;
    int * nnz_local_c_idx;
    float * nnz_values;
};

__global__ void CudaMergeResults(SubBlock * d_sbs, float * d_x, float * d_y, int nblocks, int mem_b_size, int nrows, int ncols, float * sub_y_arr) {
    // Define indices for grid-stride loop
    int blockId = blockIdx.x;
    int threadId = threadIdx.x;

    // Use grid-stride loop to distribute the work across threads
    for (int i = blockId; i < nblocks; i += gridDim.x) {
        int *outLocs = d_sbs[i].nnz_global_o_idx;
        for (int j = threadId; j < mem_b_size; j += blockDim.x) {
            atomicAdd(&d_y[outLocs[j] - 1], sub_y_arr[i * mem_b_size + j]);
        }
    }
}