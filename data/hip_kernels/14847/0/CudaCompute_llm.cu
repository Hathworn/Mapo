#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
//#define DEBUG
//#define HANDLE_ERROR(x) if((x) != 0) cout << "Error!" << endl;

using namespace std;

struct SubBlock{
    int * nnz_global_i_idx;
    int * nnz_global_o_idx;
    int nnz;
    int * nnz_local_r_idx;
    int * nnz_local_c_idx;
    float * nnz_values;
};

__global__ void CudaCompute(SubBlock * d_sbs, float * d_x, float * d_y, int nblocks, int mem_b_size, int nrows, int ncols , float * sub_y_arr) {
    int blockId = blockIdx.x;
    if (blockId >= nblocks) return;

    SubBlock * work_sb = &d_sbs[blockId];

    extern __shared__ float shared_mem[];
    float * x_sub = shared_mem;
    float * y_sub = shared_mem + mem_b_size;

    // Load x_sub
    for (int i = threadIdx.x; i < mem_b_size; i += blockDim.x) {
        int global_i_idx = work_sb->nnz_global_i_idx[i];
        x_sub[i] = (global_i_idx > 0 && global_i_idx <= ncols) ? d_x[global_i_idx - 1] : 0.0f;
    }
    __syncthreads();

    // Compute y_sub
    for (int i = threadIdx.x; i < work_sb->nnz; i += blockDim.x) {
        int x_sub_idx = work_sb->nnz_local_c_idx[i] - 1;
        int y_sub_idx = work_sb->nnz_local_r_idx[i] - 1;
        atomicAdd(&y_sub[y_sub_idx], work_sb->nnz_values[i] * x_sub[x_sub_idx]);
    }
    __syncthreads();

    // Store result in sub_y_arr
    for (int i = threadIdx.x; i < mem_b_size; i += blockDim.x) {
        sub_y_arr[blockId * mem_b_size + i] = y_sub[i];
    }
}