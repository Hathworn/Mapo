#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Optimize access by using one-dimensional block and thread index calculations.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop over elements to maximize parallelism.
    for (int pos = idx; pos < w * h; pos += stride)
    {
        image[pos] = value;
    }
}