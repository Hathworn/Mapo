#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Use shared memory for data reuse
    __shared__ float shared_x[256]; // Assuming max blockDim.x is 256
    __shared__ float shared_y[256]; // Assuming max blockDim.x is 256
    
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        short2 loc = __ldg(&loc_[ptidx]); // Use __ldg to load global memory as read-only

        // Calculate scaled values
        float scaled_x = loc.x * scale;
        float scaled_y = loc.y * scale;

        // Store in shared memory
        shared_x[threadIdx.x] = scaled_x;
        shared_y[threadIdx.x] = scaled_y;

        // Ensure all threads have written to shared memory
        __syncthreads();

        // Copy from shared memory to global memory
        x[ptidx] = shared_x[threadIdx.x];
        y[ptidx] = shared_y[threadIdx.x];
    }
}