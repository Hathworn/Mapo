#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global thread index for x and y
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Return if out of bounds
    if (i >= h || j >= w) return;

    const int pos = i * s + j;

    // Inline variable assignment to avoid intermediate create and calculate inverse in one line
    float invScale = (normalization_factor[pos] == 0.0f) ? 1.0f : (1.0f / normalization_factor[pos]);
    
    // Directly multiply scaled value
    image[pos] *= invScale;
}