#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst) {
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Early return for out of bounds threads to save computation
    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate target pixel coordinates and represent the fractional parts
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py, dx, dy;
    dx = modff (cx, &px);
    dy = modff (cy, &py);

    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];
    float weight;

    // Atomic operations combined with bounds checking to reduce redundancy
    #define SAFE_ATOMIC_ADD(ty, tx, dx, dy)      \
        if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0))) { \
            weight = dx * dy;                    \
            _atomicAdd(dst + ty * image_stride + tx, value * weight); \
            _atomicAdd(normalization_factor + ty * image_stride + tx, weight); \
        }

    SAFE_ATOMIC_ADD(ty, tx, dx, dy)
    SAFE_ATOMIC_ADD(ty, tx - 1, (1.0f - dx), dy)
    SAFE_ATOMIC_ADD(ty - 1, tx - 1, (1.0f - dx), (1.0f - dy))
    SAFE_ATOMIC_ADD(ty - 1, tx, dx, (1.0f - dy))

    // Undefine macro to prevent pollution
    #undef SAFE_ATOMIC_ADD
}