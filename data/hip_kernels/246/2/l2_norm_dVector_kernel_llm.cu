#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_norm_dVector_kernel(double *a, double *partial_sum, int n) {
    __shared__ double partial_sums[BLOCKSIZE];
    
    double local_sum = 0;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int partial_index = threadIdx.x;

    // Unroll the loop to increase instruction-level parallelism
    for (; id < n; id += (blockDim.x * gridDim.x)) {
        local_sum += (a[id] * a[id]);
    }

    partial_sums[partial_index] = local_sum;
    __syncthreads();

    // Optimize reduction using loop unrolling
    for (int sum_level = blockDim.x >> 1; sum_level > 0; sum_level >>= 1) {
        if (partial_index < sum_level) {
            partial_sums[partial_index] += partial_sums[partial_index + sum_level];
        }
        __syncthreads();
    }

    // Write result of this block's sum
    if (partial_index == 0) {
        partial_sum[blockIdx.x] = partial_sums[0];
    }
}