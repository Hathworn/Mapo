#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Using __restrict__ pointers for better memory access optimization
// Launch with minimum threads and blocks required
__global__ void add_dVector_kernel(double *__restrict__ a, double *__restrict__ b, double *__restrict__ c, int n) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Unrolling loop for potential performance boost
    if (id * 4 < n) {
        int index = id * 4;
        c[index] = a[index] + b[index];
        if (index + 1 < n) c[index + 1] = a[index + 1] + b[index + 1];
        if (index + 2 < n) c[index + 2] = a[index + 2] + b[index + 2];
        if (index + 3 < n) c[index + 3] = a[index + 3] + b[index + 3];
    }
}