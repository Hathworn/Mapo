#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCKSIZE 1024

__global__ void dotProduct_dVector_kernel(double *a, double *b, double *partial_sum, int n) {
    __shared__ double partial_sums[BLOCKSIZE];

    double local_sum = 0;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int partial_index = threadIdx.x;

    // Improve efficiency: unroll the loop for accumulating product
    while (id < n) {
        local_sum += a[id] * b[id];
        id += blockDim.x * gridDim.x;
    }

    partial_sums[partial_index] = local_sum;

    __syncthreads();

    // Use warp shuffle for reduction if possible for better performance
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (partial_index < offset) {
            partial_sums[partial_index] += partial_sums[partial_index + offset];
        }
        __syncthreads();
    }

    if (partial_index == 0) {
        partial_sum[blockIdx.x] = partial_sums[0];
    }
}