#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void simple_vbo_kernel(float4 *pos, unsigned int width, unsigned int height, float time)
{
    // Calculate global thread index
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x >= width || y >= height) return; // Boundary check to avoid out-of-bounds access

    // Calculate uv coordinates
    float u = x / (float)width;
    float v = y / (float)height;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;

    // Calculate simple sine wave pattern
    float freq = 4.0f;
    float w = __sinf(u * freq + time) * __cosf(v * freq + time) * 0.5f; // Use fast math intrinsics

    // Write output vertex
    pos[y * width + x] = make_float4(u, -0.55f, v, 1.0f);
}