#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void interleave(float* input, float* output, int size) {
    // Calculate stride up front
    const int stride = blockDim.x * gridDim.x;

    // Simplified loop condition and increment logic
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += stride) {
        output[2 * i] = input[i];
        output[2 * i + 1] = input[size + 2 + i];
    }
}