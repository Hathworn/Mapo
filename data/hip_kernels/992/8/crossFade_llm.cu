#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void crossFade(float* out1, float* out2, int numFrames) {
    // Calculate a global index for the current thread
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure threadID does not exceed numFrames
    if (threadID < numFrames) {
        // Calculate fade factor once for the current thread
        float fn = float(threadID) / (numFrames - 1.0f);
        
        // Apply crossfade for two consecutive output indices
        int idx = threadID * 2;
        out1[idx] = out1[idx] * (1.0f - fn) + out2[idx] * fn;
        out1[idx + 1] = out1[idx + 1] * (1.0f - fn) + out2[idx + 1] * fn;
    }
}