#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MyFloatScale(float *a, float scale, int size) {
    // Efficient thread index calculation
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    // Optimize memory access by using striding loop
    for (int i = threadID; i < size; i += stride) {
        a[i] *= scale;
    }
}