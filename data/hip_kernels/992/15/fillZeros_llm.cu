#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fillZeros(float* buf, int size) {
    // Calculate the unique thread index using improved readability
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use the stride to cover all elements in a single loop
    for (int i = idx; i < size; i += stride) {
        buf[i] = 0.0f; // Assign zero directly
    }
}