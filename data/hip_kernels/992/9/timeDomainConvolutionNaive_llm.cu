#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void timeDomainConvolutionNaive(float* ibuf, float* rbuf, float* obuf, long long oframes, long long rframes, int ch, float gain) {
    int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadID < oframes) { // Ensure threadID is within valid range
        float value = 0.0f;
        for (int k = 0; k < rframes; k++) {
            if (threadID >= k) { // Ensure indexing within bounds
                value += ibuf[threadID - k] * rbuf[k];
            }
        }
        obuf[threadID * 2 + ch] = value * gain;
    }
}