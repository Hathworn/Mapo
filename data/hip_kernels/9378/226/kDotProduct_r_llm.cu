#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;
    if (eidx < numElements) {
        // Unroll loop by 4 for coalesced memory access
        for (uint end = eidx + gridDim.x * DP_BLOCKSIZE * 4; eidx < min(numElements, end); eidx += gridDim.x * DP_BLOCKSIZE * 4) {
            shmem[threadIdx.x] += a[eidx] * b[eidx];
            if (eidx + gridDim.x * DP_BLOCKSIZE < numElements) shmem[threadIdx.x] += a[eidx + gridDim.x * DP_BLOCKSIZE] * b[eidx + gridDim.x * DP_BLOCKSIZE];
            if (eidx + 2 * gridDim.x * DP_BLOCKSIZE < numElements) shmem[threadIdx.x] += a[eidx + 2 * gridDim.x * DP_BLOCKSIZE] * b[eidx + 2 * gridDim.x * DP_BLOCKSIZE];
            if (eidx + 3 * gridDim.x * DP_BLOCKSIZE < numElements) shmem[threadIdx.x] += a[eidx + 3 * gridDim.x * DP_BLOCKSIZE] * b[eidx + 3 * gridDim.x * DP_BLOCKSIZE];
        }
    }
    __syncthreads();

    // Use loop unrolling for reduction
    for (uint stride = DP_BLOCKSIZE / 2; stride > 32; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x < 32) {
        volatile float* mysh = &shmem[threadIdx.x];
        *mysh += mysh[32];
        *mysh += mysh[16];
        *mysh += mysh[8];
        *mysh += mysh[4];
        *mysh += mysh[2];
        *mysh += mysh[1];
    }

    // Only first thread writes to target
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}