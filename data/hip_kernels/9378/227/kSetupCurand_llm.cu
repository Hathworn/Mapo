#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x; // Optimize calculation of thread index
    hiprand_init(seed, tidx, 0, &state[tidx]); // Each thread initializes its own state with a unique seed
}