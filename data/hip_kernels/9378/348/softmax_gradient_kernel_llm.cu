#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    // Using shared memory for reduction buffer to optimize memory access
    __shared__ float shared_buffer[SOFTMAX_NUM_THREADS];
    
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;

    const int thread_id = threadIdx.x;
    float temp_sum = 0;

    // Parallelized reduction calculation
    for (int i = thread_id; i < dim; i += blockDim.x) {
        temp_sum += dY[i] * Y[i];
    }
    shared_buffer[thread_id] = temp_sum;
    __syncthreads();

    // Use a single warp to finalize reduction which may increase efficiency
    if (thread_id < warpSize) {
        for (int offset = warpSize; offset < blockDim.x; offset += warpSize) {
            shared_buffer[thread_id] += shared_buffer[thread_id + offset];
        }
    }

    __syncthreads();

    // Compute gradient with the reduced sum
    temp_sum = shared_buffer[0];
    for (int i = thread_id; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - temp_sum);
    }
}