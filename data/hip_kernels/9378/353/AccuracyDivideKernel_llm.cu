#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Utilize thread blocks to allow parallel execution
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index
    if (idx == 0) { // Ensure only one thread performs the division
        *accuracy /= N;
    }
}