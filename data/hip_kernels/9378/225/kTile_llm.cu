#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate the global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Total number of elements in the target matrix
    const unsigned int numEls = tgtWidth * tgtHeight;

    // Optimize access by processing contiguous memory regions
    const int threadCount = blockDim.x * gridDim.x;
    for (uint i = idx; i < numEls; i += threadCount) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Fetch from source matrix and store in the target matrix
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}