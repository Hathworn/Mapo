#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define TILE_SIZE 16

__global__ void computeMoment(int8_t *readArr, int8_t *writeArr, float *weightArr, int n, int tileSize) {
    int row_init = blockIdx.x * (blockDim.x * tileSize) + threadIdx.x * tileSize;
    int col_init = blockIdx.y * (blockDim.y * tileSize) + threadIdx.y * tileSize;

    // Use shared memory for weightArr
    __shared__ float sharedWeightArr[5][5];
    if (threadIdx.x < 5 && threadIdx.y < 5) {
        sharedWeightArr[threadIdx.x][threadIdx.y] = weightArr[threadIdx.x * 5 + threadIdx.y];
    }
    __syncthreads();

    for (int ii = 0; ii < tileSize; ++ii) {
        for (int jj = 0; jj < tileSize; ++jj) {
            int row = row_init + ii;
            int col = col_init + jj;

            // If coordinates are between boundaries
            if (row < n && col < n) {
                float influence = 0.0f;
                for (int i = -2; i < 3; i++) {
                    for (int j = -2; j < 3; j++) {
                        int y = (row + i + n) % n;
                        int x = (col + j + n) % n;
                        influence += sharedWeightArr[i + 2][j + 2] * readArr[y * n + x];
                    }
                }

                writeArr[row * n + col] = readArr[row * n + col];
                if (influence < -diff) writeArr[row * n + col] = -1;
                else if (influence > diff) writeArr[row * n + col] = 1;
            }
        }
    }
}