#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeMoment(int *readArr, int *writeArr, double *weightArr, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // Efficient boundary check and thread culling
    if (row < 517 && col < 517) {
        float influence = 0.0f;

        // Unrolling loops for higher performance
        #pragma unroll
        for (int i = -2; i < 3; i++) {
            #pragma unroll
            for (int j = -2; j < 3; j++) {
                int y = (row + i + n) % n;
                int x = (col + j + n) % n;
                influence += weightArr[(i + 2) * 5 + (j + 2)] * readArr[y * n + x];
            }
        }

        int index = row * n + col;
        writeArr[index] = readArr[index];
        if (influence < -diff) 
            writeArr[index] = -1;
        else if (influence > diff) 
            writeArr[index] = 1;
    }
    // Synchronization removed as it is unnecessary here
}