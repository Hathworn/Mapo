#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeMoment(int *readArr, int *writeArr, double *weightArr, int n) {
    __shared__ int readArr_shared[32][32];
    __shared__ double weightArr_shared[5][5];

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    // Load weightArr to shared memory with thread coalescing
    if (threadIdx.x < 5 && threadIdx.y < 5) {
        weightArr_shared[threadIdx.x][threadIdx.y] = weightArr[threadIdx.x * 5 + threadIdx.y];
    }
    __syncthreads();

    // Load readArr into shared memory
    if (row < n && col < n) {
        readArr_shared[threadIdx.x][threadIdx.y] = readArr[row * n + col];
    }
    __syncthreads();

    if (row < n && col < n) {
        float influence = 0.0f;
        // Optimize boundaries check by combining conditions
        if (threadIdx.x >= MIN_MARGIN && threadIdx.y >= MIN_MARGIN &&
            threadIdx.x <= 31 - MIN_MARGIN && threadIdx.y <= 31 - MIN_MARGIN) {
            for (int i = -2; i < 3; i++) {
                for (int j = -2; j < 3; j++) {
                    influence += weightArr_shared[i + 2][j + 2] * readArr_shared[threadIdx.x + i][threadIdx.y + j];
                }
            }
            writeArr[row * n + col] = readArr_shared[threadIdx.x][threadIdx.y];
        } else {
            for (int i = -2; i < 3; i++) {
                for (int j = -2; j < 3; j++) {
                    int y = (row + i + n) % n;
                    int x = (col + j + n) % n;
                    influence += weightArr_shared[i + 2][j + 2] * readArr[y * n + x];
                }
            }
            writeArr[row * n + col] = readArr[row * n + col];
        }

        // Simplified influence assignment
        if (influence < -diff) {
            writeArr[row * n + col] = -1;
        } else if (influence > diff) {
            writeArr[row * n + col] = 1;
        }
    }
    __syncthreads();
}