#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kInitClusters(const hipSurfaceObject_t surfFrameLab, float* clusters, int width, int height, int nSpxPerRow, int nSpxPerCol) {
    // Calculate the global index for the current thread
    int centroidIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int nSpx = nSpxPerCol * nSpxPerRow;

    // Process only within the valid range
    if (centroidIdx < nSpx) {
        int wSpx = width / nSpxPerRow; // Width of each superpixel
        int hSpx = height / nSpxPerCol; // Height of each superpixel

        int i = centroidIdx / nSpxPerRow; // Row index of centroid
        int j = centroidIdx % nSpxPerRow; // Column index of centroid

        // Compute the central coordinates of the superpixel
        int x = j * wSpx + wSpx / 2;
        int y = i * hSpx + hSpx / 2;

        float4 color;
        // Efficiently read color values using surface object
        surf2Dread(&color, surfFrameLab, x * sizeof(float4), y); // Use sizeof(float4) for proper byte addressing

        // Store the color and spatial information into the cluster array
        clusters[centroidIdx] = color.x;
        clusters[centroidIdx + nSpx] = color.y;
        clusters[centroidIdx + 2 * nSpx] = color.z;
        clusters[centroidIdx + 3 * nSpx] = x;
        clusters[centroidIdx + 4 * nSpx] = y;
    }
}