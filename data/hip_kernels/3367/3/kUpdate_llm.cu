#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kUpdate(int nbSpx, float* clusters, float* accAtt_g)
{
    int cluster_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use temporary variables to store computed offsets
    if (cluster_idx < nbSpx) {
        int offset1 = cluster_idx + nbSpx;
        int offset2 = cluster_idx + 2 * nbSpx;
        int offset3 = cluster_idx + 3 * nbSpx;
        int offset4 = cluster_idx + 4 * nbSpx;
        int offset5 = cluster_idx + 5 * nbSpx;
        
        int counter = accAtt_g[offset5];
        
        if (counter != 0) {
            // Use precomputed offsets for efficient memory access
            clusters[cluster_idx] = accAtt_g[cluster_idx] / counter;
            clusters[offset1] = accAtt_g[offset1] / counter;
            clusters[offset2] = accAtt_g[offset2] / counter;
            clusters[offset3] = accAtt_g[offset3] / counter;
            clusters[offset4] = accAtt_g[offset4] / counter;

            // Reset accumulators using precomputed offsets
            accAtt_g[cluster_idx] = 0;
            accAtt_g[offset1] = 0;
            accAtt_g[offset2] = 0;
            accAtt_g[offset3] = 0;
            accAtt_g[offset4] = 0;
            accAtt_g[offset5] = 0;
        }
    }
}