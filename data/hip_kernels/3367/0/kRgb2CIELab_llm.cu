#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kRgb2CIELab(const hipTextureObject_t texFrameBGRA, hipSurfaceObject_t surfFrameLab, int width, int height) {

    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;

    if (px < width && py < height) {
        uchar4 nPixel = tex2D<uchar4>(texFrameBGRA, px, py); // Fetch pixel

        // Normalize pixel values
        float _b = (float)nPixel.x / 255.0f;
        float _g = (float)nPixel.y / 255.0f;
        float _r = (float)nPixel.z / 255.0f;

        // Convert RGB to XYZ color space
        float x = _r * 0.412453f + _g * 0.357580f + _b * 0.180423f;
        float y = _r * 0.212671f + _g * 0.715160f + _b * 0.072169f;
        float z = _r * 0.019334f + _g * 0.119193f + _b * 0.950227f;

        x /= 0.950456f;
        z /= 1.088754f;

        // Use a function to simplify the gamma correction step
        auto gammaCorrection = [](float value) {
            return (value > 0.008856f) ? cbrtf(value) : (7.787f * value + 0.13793f);
        };

        float fx = gammaCorrection(x);
        float fy = gammaCorrection(y);
        float fz = gammaCorrection(z);

        // Calculate L, a, b values
        float l = y > 0.008856f ? (116.0f * fy - 16.0f) : 903.3f * y;
        float a = (fx - fy) * 500.0f;
        float b = (fy - fz) * 200.0f;

        float4 fPixel;
        fPixel.x = l;
        fPixel.y = a;
        fPixel.z = b;
        fPixel.w = 0.0f;

        // Write output
        surf2Dwrite(fPixel, surfFrameLab, px * sizeof(float4), py);
    }
}