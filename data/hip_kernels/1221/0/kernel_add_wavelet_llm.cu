#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to add wavelet source term
__global__ void kernel_add_wavelet(float *d_u, float *d_wavelet, int it)
{
    // Calculate global index
    unsigned int idx = blockIdx.y * blockDim.y * c_nx + blockIdx.x * blockDim.x + threadIdx.y * c_nx + threadIdx.x;

    // Add wavelet to source position
    if (idx == c_jsrc * c_nx + c_isrc)
    {
        d_u[idx] += d_wavelet[it];
    }
}