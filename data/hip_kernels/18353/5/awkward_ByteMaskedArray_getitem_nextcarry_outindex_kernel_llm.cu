#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void awkward_ByteMaskedArray_getitem_nextcarry_outindex_kernel(int64_t* prefixed_mask, int64_t* to_carry, int64_t* outindex, int8_t* mask, int64_t length) {
    // Calculate unique thread index across the entire grid
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread_id is within bounds
    if(thread_id < length) {
        int64_t mask_value = prefixed_mask[thread_id];
        if (mask[thread_id] != 0) {
            // Write to to_carry and outindex for active mask
            to_carry[mask_value - 1] = thread_id;
            outindex[thread_id] = mask_value - 1;
        } else {
            // Set outindex to -1 for inactive mask
            outindex[thread_id] = -1;
        }
    }
}