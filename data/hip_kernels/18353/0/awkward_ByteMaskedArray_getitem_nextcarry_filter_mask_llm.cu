#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void awkward_ByteMaskedArray_getitem_nextcarry_filter_mask(int8_t* mask, bool validwhen, int64_t length) {
  // Calculate the global thread ID more efficiently
  int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
  
  // Ensure thread_id remains within bounds of the array length
  if(thread_id < length) {
    // Simplify the condition to a more concise form
    mask[thread_id] = ((mask[thread_id] != 0) == validwhen) ? 1 : mask[thread_id];
  }
}