#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void awkward_Content_getitem_next_missing_jagged_getmaskstartstop_kernel(
    int64_t* prefixed_index, int64_t* index_in, int64_t* offsets_in,
    int64_t* mask_out, int64_t* starts_out, int64_t* stops_out, int64_t length) {

    // Calculate global thread index.
    int64_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread_id is within bounds.
    if (thread_id < length) {
        int64_t pre_in = prefixed_index[thread_id] - 1;
        starts_out[thread_id] = offsets_in[pre_in];

        // Use branchless operations to optimize divergent branches.
        int64_t index_val = index_in[thread_id];
        int mask_value = index_val < 0 ? -1 : thread_id;
        mask_out[thread_id] = mask_value;

        int64_t stop_offset = index_val < 0 ? offsets_in[pre_in] : offsets_in[pre_in + 1];
        stops_out[thread_id] = stop_offset;
    }
}