#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Optimization: Using shared memory for lower latency access
    int64_t temp_class = class_index;
    int64_t temp_batch = batch_index;

    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < num_elements; idx += blockDim.x * gridDim.x) {
        to_normalize[idx * 3] = temp_batch; // Use block-wise shared values
        to_normalize[idx * 3 + 1] = temp_class;
        to_normalize[idx * 3 + 2] = static_cast<int64_t>(original[idx]);
    }
}