#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Setup random state with each thread handling different initial sequence and offset
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Unique thread ID
    hiprand_init(seed, id, 0, &state[id]); // Use 'id' to vary the initial sequence per thread
}