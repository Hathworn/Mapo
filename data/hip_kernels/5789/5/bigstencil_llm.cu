#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bigstencil(int* in, int* out) {
    // Cache the value of in[i] to register
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int value = in[i];
    out[i] = value + 2;
}