#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_add(float* __restrict__ a, const float* __restrict__ b, int n)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform operations within bounds
    if(i < n) {
        a[i] += b[i];
    }
}