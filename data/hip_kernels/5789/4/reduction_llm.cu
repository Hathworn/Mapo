#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];
    
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    tmp[threadIdx.x] = input[idx]; // Load input elements into shared memory
    
    __syncthreads();
    
    // Reduce in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            tmp[threadIdx.x] += tmp[threadIdx.x + s];
        }
        __syncthreads(); // Synchronize to make sure all additions are done
    }
    
    // Write result for this block to output array
    if (threadIdx.x == 0) {
        output[blockIdx.x] = tmp[0];
    }
}