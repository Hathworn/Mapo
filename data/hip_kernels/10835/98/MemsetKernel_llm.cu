#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

// Optimize memory coalescing and boundary checking
__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (row < h && col < w) { // Only proceed if within bounds
        int index = row * w + col;
        image[index] = value;
    }
}