#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate global thread index
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread index is within range
    if (ptidx < npoints)
    {
        // Enhanced memory access and computation handling
        short2 loc = loc_[ptidx];
        float scaled_x = loc.x * scale;
        float scaled_y = loc.y * scale;

        // Prevent redundant computations and improve memory coalescing
        x[ptidx] = scaled_x;
        y[ptidx] = scaled_y;
    }
}