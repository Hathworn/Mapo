#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // pixel containing bottom left corner
    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    // unify bounds checking logic
    auto bounded_add = [&](int offset, float weight) {
        if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
            _atomicAdd(dst + offset, weight * value);
            _atomicAdd(normalization_factor + offset, weight);
        }
    };

    // target pixel integer coords
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);
    float value = src[image_row_offset + j];

    // fill pixel containing bottom right corner
    int offset = ty * image_stride + tx;
    bounded_add(offset, dx * dy);

    // fill pixel containing bottom left corner
    offset--;
    bounded_add(offset, (1.0f - dx) * dy);

    // fill pixel containing upper left corner
    ty--;
    offset -= image_stride;
    bounded_add(offset, (1.0f - dx) * (1.0f - dy));

    // fill pixel containing upper right corner
    tx++;
    offset++;
    bounded_add(offset, dx * (1.0f - dy));
}