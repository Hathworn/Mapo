#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MAX 26

int b[1000]; 
unsigned long long tries = 0;
char alphabet[] = { 'a', 'b', 'c', 'd', 'e', 'f', 'g', 'h', 'i', 'j', 'k', 'l', 'm', 'n', 'o', 'p', 'q', 'r', 's', 't', 'u', 'v', 'w', 'x', 'y', 'z' };
size_t result = 1000 * sizeof(float);

int *a = (int *) malloc(result);

__global__ void parallel_passwordCrack(int length, int *d_output, int *a)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int nIter;
    __shared__ int idT;
    __shared__ long totalAttempt;

    int newB[1000] = {0};  // Initialize local array to zero
    bool cracked = false;

    if(idx == 0){
        nIter = 0;
        totalAttempt = 0;
    }
    __syncthreads();  // Ensure shared variables are initialized

    do {
        // Increment the password attempt
        newB[0]++;
        for(int i = 0; i < length; i++){
            if(newB[i] >= 26){
                newB[i] -= 26;
                newB[i+1]++;
            } else {
                break;
            }
        }

        cracked = true;
        for(int k = 0; k < length; k++) {
            if(newB[k] != a[k]){
                cracked = false;
                break;
            }
        }

        // Use __syncthreads to sync threads checking or updating shared data
        __syncthreads(); 

        if(cracked && nIter == 0) {
            idT = idx;
            break;
        } else if(nIter) {
            break;
        }

        if(idx == 0) {
            totalAttempt++;
        }

        __syncthreads();  // Ensure totalAttempt is updated

    } while(!cracked || !nIter);

    if(idx == idT) {
        for(int i = 0; i < length; i++) {
            d_output[i] = newB[i];
        }
    }
}