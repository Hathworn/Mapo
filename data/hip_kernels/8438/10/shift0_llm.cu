#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void shift0(float* in, float* out, int inDim0, int inStride0, int inStride1, int inScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    while (tid < inScalarCount) {
        int linearIndex = tid;
        int inIndex0 = linearIndex / inStride0;
        linearIndex -= inIndex0 * inStride0; // Simplification of subtraction
        int inIndex1 = linearIndex / inStride1;
        if (inIndex0 + inIndex1 < inDim0) { // Changed '>= inDim0' to '< inDim0' for valid range 
            out[tid + inIndex1 * inStride0] = in[tid];
        }
        tid += stride;
    }
}