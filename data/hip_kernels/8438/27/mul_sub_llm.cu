#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void mul_sub(float* in1, float* in2, float* out, int in1ScalarCount, int in2ScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use loop unrolling for better performance
    for (int i = tid; i < in1ScalarCount; i += stride * 4) {
        if (i < in1ScalarCount)
            out[i] = in1[i] * in2[i % in2ScalarCount];
        if (i + stride < in1ScalarCount)
            out[i + stride] = in1[i + stride] * in2[(i + stride) % in2ScalarCount];
        if (i + 2 * stride < in1ScalarCount)
            out[i + 2 * stride] = in1[i + 2 * stride] * in2[(i + 2 * stride) % in2ScalarCount];
        if (i + 3 * stride < in1ScalarCount)
            out[i + 3 * stride] = in1[i + 3 * stride] * in2[(i + 3 * stride) % in2ScalarCount];
    }
}