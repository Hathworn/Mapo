#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_sub_grad(float* in1_x, float* in1_d, float* in2_x, float* in2_d, float* out, int in1ScalarCount, int in2ScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    while (tid < in1ScalarCount) {
        int index = tid % in2ScalarCount;
        
        // Load from global memory into registers to reduce memory access latency
        float out_tid = out[tid];
        float in2_x_index = in2_x[index];
        float in1_x_tid = in1_x[tid];

        // Use register values for computations
        in1_d[tid] += out_tid * in2_x_index;
        in2_d[tid] = in1_x_tid * out_tid;

        tid += stride;
    }
}