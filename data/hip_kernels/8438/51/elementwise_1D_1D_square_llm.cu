#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_square(float* in, float* out, int size) {
    // Calculate unique global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread operates within bounds
    if (tid < size) {
        out[tid] = in[tid] * in[tid];
    }
}