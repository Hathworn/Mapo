#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void clipAt(float* in, float bound, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Unrolling the loop to improve performance
    for (; tid < size; tid += stride * 2) {
        if (in[tid] > bound) in[tid] = bound;
        else if (in[tid] < -bound) in[tid] = -bound;
        // Process the next element in parallel
        int next_tid = tid + stride;
        if (next_tid < size) {
            if (in[next_tid] > bound) in[next_tid] = bound;
            else if (in[next_tid] < -bound) in[next_tid] = -bound;
        }
    }
}