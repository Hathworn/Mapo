#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_add(float* in1, float* in2, float* out, int size) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single if condition for range check to reduce branch divergence
    if (tid < size) {
        out[tid] = in1[tid] + in2[tid];

        // Loop through remaining elements using grid stride
        tid += gridDim.x * blockDim.x;
        while (tid < size) {
            out[tid] = in1[tid] + in2[tid];
            tid += gridDim.x * blockDim.x;
        }
    }
}