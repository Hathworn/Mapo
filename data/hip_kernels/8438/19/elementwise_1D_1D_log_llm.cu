#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_log(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop through increments of stride starting at tid
    for (int idx = tid; idx < size; idx += stride) {
        out[idx] = logf(in[idx]); // Use logf for single precision
    }
}