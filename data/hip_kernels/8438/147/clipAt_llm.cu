#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clipAt(float* in, float bound, int size) {
    // Using stride-based iteration for better memory access.
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    while (tid < size) {
        // Combined condition checks for better performance.
        if (in[tid] > bound) {
            in[tid] = bound;
        } else if (in[tid] < -bound) {
            in[tid] = -bound;
        }
        tid += stride;
    }
}