#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void arrayFill_greg(float* data, float value, int size) {
    // Calculate the global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use a more efficient loop unrolling technique
    for (int i = tid; i < size; i += stride * 4) {
        data[i] = value;
        if (i + stride < size) data[i + stride] = value;
        if (i + 2 * stride < size) data[i + 2 * stride] = value;
        if (i + 3 * stride < size) data[i + 3 * stride] = value;
    }
}