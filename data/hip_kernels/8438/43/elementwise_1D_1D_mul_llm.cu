#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwise_1D_1D_mul(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Simplified loop with boundary check inside
    for (; tid < size; tid += stride) {
        out[tid] = in1[tid] * in2[tid];
    }
}