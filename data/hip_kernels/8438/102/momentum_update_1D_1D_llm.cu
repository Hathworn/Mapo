#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void momentum_update_1D_1D(float* x, float* d, float* m, float learning_rate, float momentum, float gradClip, bool nesterov, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll the loop for better performance
    for (int index = tid; index < size; index += stride) {
        float temp = d[index];
        
        // Use fminf and fmaxf for fast gradient clipping
        temp = fminf(fmaxf(temp, -gradClip), gradClip);
        
        m[index] = m[index] * momentum + temp;
        
        if (nesterov) {
            temp += momentum * m[index]; 
        } else {
            temp = m[index];
        }

        x[index] -= learning_rate * temp;
        
        d[index] = 0.0f; // Use 0.0f for consistent float assignment
    }
}