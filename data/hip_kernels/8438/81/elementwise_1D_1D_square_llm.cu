#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwise_1D_1D_square(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop to handle out-of-bounds within the condition
    for (; tid < size; tid += stride) {
        out[tid] = in[tid] * in[tid];
    }
}