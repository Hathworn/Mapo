#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void adagrad_update_1D_1D(float* x, float* d, float* m, float clip, float lr, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Load elements into shared memory for better performance
    __shared__ float shared_x[256]; // Assuming max blockDim.x = 256
    __shared__ float shared_d[256];
    __shared__ float shared_m[256];

    while (tid < size) {
        // Load data from global memory
        shared_x[threadIdx.x] = x[tid];
        shared_d[threadIdx.x] = d[tid];
        shared_m[threadIdx.x] = m[tid];

        __syncthreads(); // Ensure all loads are complete

        // Clip gradient
        if (shared_d[threadIdx.x] > clip) shared_d[threadIdx.x] = clip;
        if (shared_d[threadIdx.x] < -clip) shared_d[threadIdx.x] = -clip;

        // Update momentum and variable
        shared_m[threadIdx.x] += shared_d[threadIdx.x] * shared_d[threadIdx.x];
        shared_x[threadIdx.x] -= lr * shared_d[threadIdx.x] / sqrt(shared_m[threadIdx.x] + 0.00000001);

        // Reset gradient
        shared_d[threadIdx.x] = 0.0f;

        __syncthreads(); // Ensure all updates are complete

        // Write data back to global memory
        x[tid] = shared_x[threadIdx.x];
        m[tid] = shared_m[threadIdx.x];
        d[tid] = shared_d[threadIdx.x];

        tid += stride;
    }
}