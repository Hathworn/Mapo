#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_exp(float* in, float* out, int size) {
    // Calculate the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll the loop for better performance
    for (; tid < size; tid += stride * 4) {
        if (tid < size) out[tid] = exp(in[tid]);
        if (tid + stride < size) out[tid + stride] = exp(in[tid + stride]);
        if (tid + 2 * stride < size) out[tid + 2 * stride] = exp(in[tid + 2 * stride]);
        if (tid + 3 * stride < size) out[tid + 3 * stride] = exp(in[tid + 3 * stride]);
    }
}