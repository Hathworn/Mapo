#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_log(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll loop for better performance
    for (; tid < size; tid += stride * 4) {
        if (tid < size) out[tid] = logf(in[tid]);
        
        if (tid + stride < size) out[tid + stride] = logf(in[tid + stride]);
        
        if (tid + 2 * stride < size) out[tid + 2 * stride] = logf(in[tid + 2 * stride]);
        
        if (tid + 3 * stride < size) out[tid + 3 * stride] = logf(in[tid + 3 * stride]);
    }
}