#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_sub(float* in1, float* in2, float* out, int in1ScalarCount, int in2ScalarCount) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Eliminate stride-based loop by leveraging kernel launch configuration
    if (tid < in1ScalarCount) {
        // Perform multiplication and subtraction
        out[tid] = in1[tid] * in2[tid % in2ScalarCount];
    }
}