#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void nllLoss_grad(int x_stride, float *yGrad, int* target, float* xGrad) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure tid is within valid range
    if (tid < gridDim.x * blockDim.x) {
        int offset = tid * x_stride + target[tid];

        // Use atomic operation to prevent race conditions
        atomicAdd(&xGrad[offset], -yGrad[tid]);
    }
}