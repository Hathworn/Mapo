#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shift0(float* in, float* out, int inDim0, int inStride0, int inStride1, int inScalarCount) {
    // Calculate thread id outside the loop
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    while (tid < inScalarCount) {
        int linearIndex = tid;
        int inIndex0 = linearIndex / inStride0;
        linearIndex -= inIndex0 * inStride0;
        int inIndex1 = linearIndex / inStride1;

        // Avoid redundant calculation of return condition
        if (inIndex0 + inIndex1 < inDim0) {
            out[tid + inIndex1 * inStride0] = in[tid];
        }
        tid += stride; // Increment tid outside loop header
    }
}