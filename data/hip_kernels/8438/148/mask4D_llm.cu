#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask4D(float* in, int* mask, int xstrides0, int xstrides1, int xstrides2, int xstrides3, int scalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use while loop for potential better optimization by certain compilers
    while (tid < scalarCount) {
        int linearIndex = tid;
        int xindex0 = linearIndex / xstrides0;
        linearIndex = linearIndex % xstrides0; // Use modulo operator for clearer intent
        int xindex1 = linearIndex / xstrides1;
        linearIndex = linearIndex % xstrides1; // Use modulo operator for clearer intent
        int xindex2 = linearIndex / xstrides2;
        int xindex3 = linearIndex % xstrides2; // Use modulo operator for clearer intent
        
        // Use temporary variable to reduce repeated memory access
        int mask_val = mask[xindex0];
        if (xindex3 >= mask_val) {
            in[tid] = 0;
        }
        
        tid += stride;
    }
}