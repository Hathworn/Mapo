#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_square_grad(float* in_x, float* in_d, float* out_x, float* out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use shared memory for coalesced access, increases data access speed
    extern __shared__ float shared_out_d[];

    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_out_d[i] = out_d[i];
    }
    __syncthreads();

    for (; tid < size; tid += stride) {
        // Ensure tid < size is checked once outside
        in_d[tid] += shared_out_d[tid] * 2.0f * in_x[tid];
    }
}