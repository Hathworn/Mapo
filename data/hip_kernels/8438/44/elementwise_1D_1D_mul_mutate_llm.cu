#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_mul_mutate(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use shared memory to optimize memory access
    extern __shared__ float sharedIn1[];
    extern __shared__ float sharedIn2[];

    if (tid < size) {
        sharedIn1[threadIdx.x] = in1[tid];
        sharedIn2[threadIdx.x] = in2[tid];
    }
    __syncthreads();

    for (; tid < size; tid += stride) {
        if (tid < size) {
            // Update output using shared memory
            out[tid] += sharedIn1[threadIdx.x] * sharedIn2[threadIdx.x];
        }
    }
}