#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize workload distribution and memory access
__global__ void elementwise_1D_1D_mul(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll loop for better performance
    for (int i = tid; i < size; i += stride) {
        out[i] = in1[i] * in2[i];
    }
}