#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwise_1D_1D_log(float* __restrict__ in, float* __restrict__ out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (; tid < size; tid += stride) {
        // Ensure thread accesses in-bounds element of input and output
        if (tid < size) {
            out[tid] = logf(in[tid]); // Use logf for single-precision floats
        }
    }
}