#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void repeat0(float* in, float* out, int outStride0, int outStride1, int outScalarCount) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the total number of threads
    int stride = gridDim.x * blockDim.x;
    
    // Use shared memory to optimize repeated global memory access
    __shared__ int shared_outStride0;
    __shared__ int shared_outStride1;
    
    // Load values into shared memory
    if (threadIdx.x == 0) {
        shared_outStride0 = outStride0;
        shared_outStride1 = outStride1;
    }
    __syncthreads();

    // Iterate over output scalars and compute input indices
    for (; tid < outScalarCount; tid += stride) {
        int linearIndex = tid;
        int outIndex0 = linearIndex / shared_outStride0;
        linearIndex = linearIndex - outIndex0 * shared_outStride0;
        int outIndex1 = linearIndex / shared_outStride1;
        int outIndex2 = linearIndex - outIndex1 * shared_outStride1;
        int inIndex = outIndex2 + (outIndex0 + outIndex1) * shared_outStride1;
        // Copy input data to output
        out[tid] = in[inIndex];
    }
}