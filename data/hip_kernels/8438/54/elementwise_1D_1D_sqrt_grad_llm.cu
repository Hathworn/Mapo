#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_sqrt_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
    // Calculate the global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Optimized loop with bounds checking
    while (tid < size) {
        in_d[tid] += out_d[tid] / out_x[tid] / 2;  // Direct arithmetic operation
        tid += stride;
    }
}