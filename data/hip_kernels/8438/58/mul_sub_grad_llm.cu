#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_sub_grad(float* in1_x, float* in1_d, float* in2_x, float* in2_d, float* out, int in1ScalarCount, int in2ScalarCount) {
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    while (tid < in1ScalarCount) {
        int index = tid % in2ScalarCount;
        
        // Perform operations using temp variables
        float out_val = out[tid];
        float in2_x_val = in2_x[index];
        in1_d[tid] += out_val * in2_x_val;
        in2_d[tid] = in1_x[tid] * out_val;
        
        // Increment thread index by stride
        tid += stride;
    }
}