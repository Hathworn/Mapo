#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_log(float* in, float* out, int size) {
    // Calculate global thread index 
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Loop unrolling for better performance
    while (tid < size) {
        out[tid] = log(in[tid]);
        tid += stride;
    }
}