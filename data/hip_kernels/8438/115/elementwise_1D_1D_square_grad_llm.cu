#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_square_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use a single loop with thread-safe atomicAdd
    for (int i = tid; i < size; i += stride) {
        atomicAdd(&in_d[i], out_d[i] * 2 * in_x[i]);
    }
}