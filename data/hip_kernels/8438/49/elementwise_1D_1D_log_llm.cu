#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_log(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll the loop to improve performance
    for (; tid < size; tid += stride * 4) {
        if (tid < size) out[tid] = log(in[tid]);
        if (tid + stride < size) out[tid + stride] = log(in[tid + stride]);
        if (tid + 2 * stride < size) out[tid + 2 * stride] = log(in[tid + 2 * stride]);
        if (tid + 3 * stride < size) out[tid + 3 * stride] = log(in[tid + 3 * stride]);
    }
}