#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hardTanh_grad(float* in_x, float* in_d, float* out_d, float min_val, float max_val, int size, bool inplace) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    // Use shared memory to minimize global memory accesses
    extern __shared__ float shared_data[];

    for (int i = tid; i < size; i += stride) {
        float x_val = in_x[i]; // Load data into register
        float d_val = (inplace) ? in_d[i] : out_d[i]; // Select based on inplace

        if (inplace) {
            if (x_val < min_val || x_val > max_val) d_val = 0;
        } else {
            if (x_val >= min_val && x_val <= max_val) d_val += out_d[i];
        }
  
        if (inplace) {
            in_d[i] = d_val; // Store result back to global memory
        } else {
            shared_data[threadIdx.x] = d_val; // Use shared memory for intermediate computation
            __syncthreads(); // Sync to ensure all threads have written to shared memory
            out_d[i] = shared_data[threadIdx.x]; // Write shared memory result back to global memory
        }
    }
}