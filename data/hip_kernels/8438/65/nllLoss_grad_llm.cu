#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void nllLoss_grad(int x_stride, float *yGrad, int* target, float* xGrad) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the unique thread index
    
    // Check if tid is within bounds to avoid illegal memory access
    if (tid < gridDim.x * blockDim.x) {
        int offset = tid * x_stride + target[tid];   // Compute the offset
        atomicAdd(&xGrad[offset], -yGrad[tid]);      // Use atomicAdd for safe concurrent updates
    }
}