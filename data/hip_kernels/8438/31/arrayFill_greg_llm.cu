#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void arrayFill_greg(float* data, float value, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Calculate stride using total threads in the grid
    int stride = gridDim.x * blockDim.x;

    // Use unrolling for increased performance
    for (int i = tid; i < size; i += stride * 4) {
        data[i] = value;
        if (i + stride < size) data[i + stride] = value;
        if (i + 2 * stride < size) data[i + 2 * stride] = value;
        if (i + 3 * stride < size) data[i + 3 * stride] = value;
    }
}