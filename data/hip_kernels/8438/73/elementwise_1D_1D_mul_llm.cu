#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_mul(float* in1, float* in2, float* out, int size) {
    // Calculate thread id
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure tid does not exceed size boundary
    if (tid < size) {
        out[tid] = in1[tid] * in2[tid];
    }
}
```
