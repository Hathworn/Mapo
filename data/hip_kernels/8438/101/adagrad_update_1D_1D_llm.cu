#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adagrad_update_1D_1D(float* x, float* d, float* m, float clip, float lr, int size) {
    // Calculate unique thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a loop with stride for processing large data
    for (int idx = tid; idx < size; idx += gridDim.x * blockDim.x) {
        // Clamp gradient values
        float grad = d[idx];
        if (grad > clip) grad = clip;
        if (grad < -clip) grad = -clip;

        // Accumulate squared gradients
        m[idx] += grad * grad;
        // Update parameter
        x[idx] -= lr * grad / sqrt(m[idx] + 1e-8f);
        
        // Reset gradient
        d[idx] = 0.0f;
    }
}