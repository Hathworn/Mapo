#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_grad(float* in, int inSize0, int inSize1, int inSize2, int inSize3, int nElement, float* out, int outStride0, int outStride1, int outStride2, int dim) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Precompute shifts for efficiency
    int inSize2_inSize3 = inSize2 * inSize3;
    int inSize1_inSize2 = inSize1 * inSize2;
    
    for (int i = tid; i < nElement; i += stride) {
        // Simplified index calculations
        int inOff1 = i / inSize2_inSize3;
        int inDim1 = inOff1 % inSize1;
        int inDim0 = inOff1 / inSize1;
        int inOff2 = i % inSize2_inSize3;
        int inDim2 = inOff2 / inSize3;
        int inDim3 = inOff2 % inSize3;

        int outOff = 0;
        // Efficient index conditional processing
        if (dim == 0)      outOff = inDim1 * outStride0 + inDim2 * outStride1 + inDim3 * outStride2;
        else if (dim == 1) outOff = inDim0 * outStride0 + inDim2 * outStride1 + inDim3 * outStride2;
        else if (dim == 2) outOff = inDim0 * outStride0 + inDim1 * outStride1 + inDim3 * outStride2;
        else if (dim == 3) outOff = inDim0 * outStride0 + inDim1 * outStride1 + inDim2 * outStride2;

        in[i] += out[outOff];
    }
}