#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nllLoss_grad(int x_stride, float *yGrad, int* target, float* xGrad) {
    // Use shared memory to cache frequently accessed data
    extern __shared__ int sharedTarget[]; 
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (threadIdx.x < x_stride) {
        sharedTarget[threadIdx.x] = target[tid];
    }
    __syncthreads();
    
    int offset = tid * x_stride + sharedTarget[threadIdx.x];
    atomicAdd(&xGrad[offset], -yGrad[tid]);  // Atomic operation to ensure correct updates
}