#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_exp(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (tid < size) { // Check if thread index is within bounds
        out[tid] = exp(in[tid]); // Perform computation if within bounds
    }
}