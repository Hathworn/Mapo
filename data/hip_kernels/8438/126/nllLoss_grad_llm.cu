#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nllLoss_grad(int x_stride, float *yGrad, int* target, float* xGrad) {
    // Calculate global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Early exit if tid exceeds dataset
    if (tid >= gridDim.x * blockDim.x) return;

    // Compute the offset and perform gradient update
    int offset = tid * x_stride + target[tid];
    atomicAdd(&xGrad[offset], -yGrad[tid]); // Use atomic operation to avoid race conditions
}