#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_sub_grad(float* in1_x, float* in1_d, float* in2_x, float* in2_d, float* out, int in1ScalarCount, int in2ScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Loop unrolling for better performance
    for (; tid < in1ScalarCount; tid += stride*4) {
        int index1 = tid % in2ScalarCount;
        in1_d[tid] += out[tid] * in2_x[index1];
        in2_d[tid] = in1_x[tid] * out[tid];
        
        // Unroll next iterations
        if (tid + stride < in1ScalarCount) {
            int index2 = (tid + stride) % in2ScalarCount;
            in1_d[tid + stride] += out[tid + stride] * in2_x[index2];
            in2_d[tid + stride] = in1_x[tid + stride] * out[tid + stride];
        }
        
        if (tid + 2*stride < in1ScalarCount) {
            int index3 = (tid + 2*stride) % in2ScalarCount;
            in1_d[tid + 2*stride] += out[tid + 2*stride] * in2_x[index3];
            in2_d[tid + 2*stride] = in1_x[tid + 2*stride] * out[tid + 2*stride];
        }
        
        if (tid + 3*stride < in1ScalarCount) {
            int index4 = (tid + 3*stride) % in2ScalarCount;
            in1_d[tid + 3*stride] += out[tid + 3*stride] * in2_x[index4];
            in2_d[tid + 3*stride] = in1_x[tid + 3*stride] * out[tid + 3*stride];
        }
    }
}