#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_square_grad(float* in_x, float* in_d, float* out_x, float* out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID

    if (tid < size) { // Ensure thread works on valid element
        for (int i = tid; i < size; i += blockDim.x * gridDim.x) { // Stride loop for threads to cover full data
            in_d[i] += out_d[i] * 2 * in_x[i]; // Efficient computation
        }
    }
}