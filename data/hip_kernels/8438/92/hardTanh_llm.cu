#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hardTanh(float* in, float* out, float min_val, float max_val, int size) {
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int stride = gridDim.x * blockDim.x;

    // Use shared memory to optimize memory access
    extern __shared__ float shared_in[];

    for (int i = tid; i < size; i += stride) {
        shared_in[threadIdx.x] = in[i];
        __syncthreads();

        // Perform computation using shared memory
        const float val = threadIdx.x < blockDim.x ? shared_in[threadIdx.x] : in[i];
        out[i] = val < min_val ? min_val : (val > max_val ? max_val : val);
    }
}