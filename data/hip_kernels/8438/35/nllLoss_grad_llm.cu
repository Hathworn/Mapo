#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nllLoss_grad(int x_stride, float *yGrad, int* target, float* xGrad) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread ID
    int offset = tid * x_stride + target[tid];        // Calculate unique offset for each thread
    atomicAdd(&xGrad[offset], -yGrad[tid]);           // Use atomic addition to prevent race conditions
}