#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_exp(float* in, float* out, int size) {
    // Calculate the thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a while loop for improved readability & efficiency
    while (tid < size) {
        out[tid] = exp(in[tid]); // Perform the element-wise exponential operation
        tid += gridDim.x * blockDim.x; // Increment to the next stride
    }
}