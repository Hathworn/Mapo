#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hardTanh_grad(float* in_x, float* in_d, float* out_d, float min_val, float max_val, int size, bool inplace) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    for (int i = tid; i < size; i += stride) {
        float x = in_x[i];  // Load memory once

        if (inplace) {
            // Set in_d[i] to zero when out of range
            in_d[i] = (x < min_val || x > max_val) ? 0 : in_d[i];
        } else {
            // Update in_d[i] only when x is within bounds
            if (x >= min_val && x <= max_val) {
                in_d[i] += out_d[i];
            }
        }
    }
}