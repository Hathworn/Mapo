#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void nllLoss(float *x, int x_stride, float *y, int* target) {
    // Calculate the unique thread ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Calculate the offset for input data
    int offset = tid * x_stride + target[tid];

    // Avoid out-of-bounds memory access
    if (tid < gridDim.x * blockDim.x) {
        y[tid] = -1.0f * x[offset]; // Use float literal for multiplication
    }
}