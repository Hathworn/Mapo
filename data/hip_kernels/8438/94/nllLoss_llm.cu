#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nllLoss(float *x, int x_stride, float *y, int* target) {
    // Precompute tid and offset only when it's within range
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < gridDim.x * blockDim.x) {
        int offset = tid * x_stride + target[tid];
        y[tid] = -x[offset]; // Use direct multiplication for -1
    }
}