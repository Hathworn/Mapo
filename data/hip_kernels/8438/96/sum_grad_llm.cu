#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_grad(float* in, int inSize0, int inSize1, int inSize2, int inSize3, int nElement, float* out, int outStride0, int outStride1, int outStride2, int dim) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Calculate index offsets outside loop for performance improvement
    int inOff2, inDim3, inOff1, inDim2, inDim0, inDim1, outOff;

    for (int i = tid; i < nElement; i += stride) {
        inOff2 = i / inSize3;
        inDim3 = i - inOff2 * inSize3;
        inOff1 = inOff2 / inSize2;
        inDim2 = inOff2 - inOff1 * inSize2;
        inDim0 = inOff1 / inSize1;
        inDim1 = inOff1 - inDim0 * inSize1;

        // Use switch-case for clarity and potential performance optimization
        switch(dim) {
            case 0:
                outOff = inDim1 * outStride0 + inDim2 * outStride1 + inDim3 * outStride2;
                break;
            case 1:
                outOff = inDim0 * outStride0 + inDim2 * outStride1 + inDim3 * outStride2;
                break;
            case 2:
                outOff = inDim0 * outStride0 + inDim1 * outStride1 + inDim3 * outStride2;
                break;
            case 3:
                outOff = inDim0 * outStride0 + inDim1 * outStride1 + inDim2 * outStride2;
                break;
        }

        in[i] += out[outOff];
    }
}