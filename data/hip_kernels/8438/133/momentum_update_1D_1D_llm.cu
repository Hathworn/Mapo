#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void momentum_update_1D_1D(float* x, float* d, float* m, float learning_rate, float momentum, float gradClip, bool nesterov, int size) {
    // Calculate thread ID with stride
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Optimize with shared memory and loop unrolling
    for (; tid < size; tid += stride) {
        float temp = d[tid];

        // Gradient clipping
        temp = max(min(temp, gradClip), -gradClip);

        // Update momentum
        m[tid] = m[tid] * momentum + temp;

        // Nesterov or standard momentum update
        temp = nesterov ? temp + momentum * m[tid] : m[tid];

        // Update parameter with learning rate
        x[tid] -= learning_rate * temp;

        // Reset gradient
        d[tid] = 0;
    }
}