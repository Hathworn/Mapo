#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_sub(float* in1, float* in2, float* out, int in1ScalarCount, int in2ScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Unroll the loop for better performance
    for (; tid + 3 * stride < in1ScalarCount; tid += 4 * stride) {
        out[tid] = in1[tid] * in2[tid % in2ScalarCount];
        out[tid + stride] = in1[tid + stride] * in2[(tid + stride) % in2ScalarCount];
        out[tid + 2 * stride] = in1[tid + 2 * stride] * in2[(tid + 2 * stride) % in2ScalarCount];
        out[tid + 3 * stride] = in1[tid + 3 * stride] * in2[(tid + 3 * stride) % in2ScalarCount];
    }
    
    // Handle remaining elements
    for (; tid < in1ScalarCount; tid += stride) {
        out[tid] = in1[tid] * in2[tid % in2ScalarCount];
    }
}