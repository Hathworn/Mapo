#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void mask4D(float* in, int* mask, int xstrides0, int xstrides1, int xstrides2, int xstrides3, int scalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use shared memory to store mask values for faster access
    __shared__ int sharedMask[256];
    
    for (int index = threadIdx.x; index < xstrides0; index += blockDim.x) {
        sharedMask[index] = mask[index];
    }
    __syncthreads();
    
    while (tid < scalarCount) {
        int linearIndex = tid;
        int xindex0 = linearIndex / xstrides0;
        linearIndex %= xstrides0;
        int xindex1 = linearIndex / xstrides1;
        linearIndex %= xstrides1;
        int xindex2 = linearIndex / xstrides2;
        int xindex3 = linearIndex % xstrides2;
        
        if (xindex3 >= sharedMask[xindex0]) in[tid] = 0;
        
        tid += stride;
    }
}