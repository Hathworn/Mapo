#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_sub(float* in1, float* in2, float* out, int in1ScalarCount, int in2ScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    int in2Idx = tid % in2ScalarCount; // Initialize index for in2 to avoid recalculating
    for (; tid < in1ScalarCount; tid += stride, in2Idx = tid % in2ScalarCount) {
        out[tid] = in1[tid] * in2[in2Idx];
    }
}