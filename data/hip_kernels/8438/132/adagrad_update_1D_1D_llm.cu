#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adagrad_update_1D_1D(float* x, float* d, float* m, float clip, float lr, int size) {
    // Calculate the thread ID and stride
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use shared memory for frequently accessed variables
    extern __shared__ float shared_memory[];

    for (; tid < size; tid += stride) {
        float d_val = d[tid];
        
        // Clip gradient
        if (d_val > clip) d_val = clip;
        if (d_val < -clip) d_val = -clip;
    
        float m_val = m[tid] + d_val * d_val;

        // Update weight
        x[tid] -= lr * d_val / sqrt(m_val + 1e-8);

        // Zero out the gradient
        d[tid] = 0;

        // Update shared memory
        shared_memory[threadIdx.x] = m_val;
        
        // Synchronize threads
        __syncthreads();

        // Write back to global memory
        m[tid] = shared_memory[threadIdx.x];
    }
}