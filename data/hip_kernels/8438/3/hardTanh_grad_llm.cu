#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void hardTanh_grad(float* in_x, float* in_d, float* out_d, float min_val, float max_val, int size, bool inplace) {
    // Calculate global thread ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    // Efficient loop unrolling with step count
    for (int i = tid; i < size; i += stride) {
        float x_val = in_x[i];
        if (inplace) {
            // Zero gradient if outside the range in inplace mode
            if (x_val < min_val || x_val > max_val) {
                in_d[i] = 0;
            }
        } else {
            // Accumulate gradient within range
            if (x_val >= min_val && x_val <= max_val) {
                in_d[i] += out_d[i];
            }
        }
    }
}