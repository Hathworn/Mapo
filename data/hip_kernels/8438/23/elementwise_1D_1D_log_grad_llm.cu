#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_log_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use a while loop for better readability and efficiency
    while (tid < size) {
        in_d[tid] += out_d[tid] / in_x[tid];
        tid += stride;
    }
}