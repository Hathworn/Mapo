#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwise_1D_1D_sqrt(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Loop unrolling for potential performance improvement
    for (; tid < size; tid += stride) {
        if (tid < size) out[tid] = sqrt(in[tid]);
        tid += stride;
        if (tid < size) out[tid] = sqrt(in[tid]);
        tid += stride;
        if (tid < size) out[tid] = sqrt(in[tid]);
        tid += stride;
        if (tid < size) out[tid] = sqrt(in[tid]);
    }
}