#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_mul_mutate(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Optimize loop by ensuring all threads are active
    while (tid < size) {
        out[tid] += in1[tid] * in2[tid];
        tid += stride;
    }
}