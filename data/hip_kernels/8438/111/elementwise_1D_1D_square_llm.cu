#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_square(float* in, float* out, int size) {
    // Use shared memory to reduce global memory access latency
    extern __shared__ float shared_in[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < size; i += stride) {
        // Load data from global to shared memory
        if (i < size)
            shared_in[threadIdx.x] = in[i];

        // Sync threads to ensure shared memory is loaded
        __syncthreads();

        // Perform computation using shared memory
        if (i < size)
            out[i] = shared_in[threadIdx.x] * shared_in[threadIdx.x];

        // Sync threads to ensure writing is done before next iteration
        __syncthreads();
    }
}