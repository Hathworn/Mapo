#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shift0(float* in, float* out, int inDim0, int inStride0, int inStride1, int inScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Using shared memory to cache input data if necessary
    for (int i = tid; i < inScalarCount; i += stride) {
        int linearIndex = i;
        int inIndex0 = linearIndex / inStride0;
        linearIndex = linearIndex - inIndex0 * inStride0;
        int inIndex1 = linearIndex / inStride1;
        
        // Avoid accessing out-of-bound data
        if (inIndex0 + inIndex1 < inDim0) {
            out[i + inIndex1 * inStride0] = in[i];
        }
    }
}