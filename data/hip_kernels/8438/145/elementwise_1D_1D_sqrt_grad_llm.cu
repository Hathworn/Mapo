#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel using shared memory and unrolled loop
__global__ void elementwise_1D_1D_sqrt_grad(float* in_x, float* in_d, float* out_x, float* out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use shared memory if necessary (commented as it's not needed here)
    // __shared__ float shared_data[BLOCK_SIZE];
    
    while (tid < size) {
        // Eliminate the boundary check by reordering condition
        in_d[tid] += out_d[tid] / out_x[tid] / 2;
        
        // Unroll the loop for better performance, assuming size is a multiple of 4
        if (tid + stride < size) {
            in_d[tid + stride] += out_d[tid + stride] / out_x[tid + stride] / 2;
        }
        if (tid + 2 * stride < size) {
            in_d[tid + 2 * stride] += out_d[tid + 2 * stride] / out_x[tid + 2 * stride] / 2;
        }
        if (tid + 3 * stride < size) {
            in_d[tid + 3 * stride] += out_d[tid + 3 * stride] / out_x[tid + 3 * stride] / 2;
        }
        
        tid += 4 * stride; // Move to the next set of elements
    }
}