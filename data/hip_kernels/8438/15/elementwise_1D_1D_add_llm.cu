#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_add(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Unroll the loop to improve performance
    for (; tid < size; tid += stride * 4) {
        if (tid < size) out[tid] = in1[tid] + in2[tid];
        if (tid + stride < size) out[tid + stride] = in1[tid + stride] + in2[tid + stride];
        if (tid + 2 * stride < size) out[tid + 2 * stride] = in1[tid + 2 * stride] + in2[tid + 2 * stride];
        if (tid + 3 * stride < size) out[tid + 3 * stride] = in1[tid + 3 * stride] + in2[tid + 3 * stride];
    }
}