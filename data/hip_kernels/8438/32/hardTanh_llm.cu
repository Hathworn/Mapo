#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hardTanh(float* in, float* out, float min_val, float max_val, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    // Unrolling loop for improved memory throughput
    for (int i = tid; i < size; i += stride * 4) {
        if (i < size) out[i] = in[i] < min_val ? min_val : (in[i] > max_val ? max_val : in[i]);
        if (i + stride < size) out[i + stride] = in[i + stride] < min_val ? min_val : (in[i + stride] > max_val ? max_val : in[i + stride]);
        if (i + 2 * stride < size) out[i + 2 * stride] = in[i + 2 * stride] < min_val ? min_val : (in[i + 2 * stride] > max_val ? max_val : in[i + 2 * stride]);
        if (i + 3 * stride < size) out[i + 3 * stride] = in[i + 3 * stride] < min_val ? min_val : (in[i + 3 * stride] > max_val ? max_val : in[i + 3 * stride]);
    }
}