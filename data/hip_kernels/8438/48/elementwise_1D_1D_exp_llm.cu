#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_exp(float* __restrict__ in, float* __restrict__ out, int size) {
    // Calculate a unique thread index optimized with memory coalescing.
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop unrolling for better performance.
    for(int i = tid; i < size; i += stride) {
        out[i] = exp(in[i]);
    }
}