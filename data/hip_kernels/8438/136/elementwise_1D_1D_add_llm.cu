#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_add(float* in1, float* in2, float* out, int size) {
    // Use shared memory to enhance memory access performance
    extern __shared__ float shared_in1[];
    extern __shared__ float shared_in2[];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input data into shared memory
    if (tid < size) {
        shared_in1[threadIdx.x] = in1[tid];
        shared_in2[threadIdx.x] = in2[tid];
    }

    __syncthreads();

    // Improved stride calculation
    int stride = blockDim.x * gridDim.x;

    for (; tid < size; tid += stride) {
        if (tid < size) {
            // Perform computation using shared memory
            out[tid] = shared_in1[threadIdx.x] + shared_in2[threadIdx.x];
        }
    }
}