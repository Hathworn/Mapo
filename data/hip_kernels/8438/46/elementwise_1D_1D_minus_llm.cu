#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_minus(float* in1, float* in2, float* out, int size) {
    // Initialize thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate stride for iteration
    int stride = blockDim.x * gridDim.x;

    // Loop over elements with stride larger than the block size
    for (; tid < size; tid += stride) {
        // Perform element-wise subtraction
        out[tid] = in1[tid] - in2[tid];
    }
}