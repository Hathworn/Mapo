#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void momentum_update_1D_1D(float* x, float* d, float* m, float learning_rate, float momentum, float gradClip, bool nesterov, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use a while loop for better readability
    while (tid < size) {
        float temp = d[tid];

        // Apply gradient clipping
        if (temp > gradClip) temp = gradClip;
        if (temp < -gradClip) temp = -gradClip;

        // Update momentum
        m[tid] = momentum * m[tid] + temp;

        // Compute the temp value based on the nesterov setting
        if (nesterov) {
            temp += momentum * m[tid];
        } else {
            temp = m[tid];
        }

        // Update x and reset d
        x[tid] -= learning_rate * temp;
        d[tid] = 0;

        tid += stride;
    }
}