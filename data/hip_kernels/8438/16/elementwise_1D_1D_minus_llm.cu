#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_minus(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use grid-stride loop for better global memory access pattern
    for (int i = tid; i < size; i += stride) {
        out[i] = in1[i] - in2[i];
    }
}