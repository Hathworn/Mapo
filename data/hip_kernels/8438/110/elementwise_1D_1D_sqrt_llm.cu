#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwise_1D_1D_sqrt(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Unroll loop for better performance
    for (int i = tid; i < size; i += stride * 4) {
        if (i < size) out[i] = sqrt(in[i]);
        if (i + stride < size) out[i + stride] = sqrt(in[i + stride]);
        if (i + 2 * stride < size) out[i + 2 * stride] = sqrt(in[i + 2 * stride]);
        if (i + 3 * stride < size) out[i + 3 * stride] = sqrt(in[i + 3 * stride]);
    }
}