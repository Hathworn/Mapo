#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hardTanh(float* in, float* out, float min_val, float max_val, int size) {
    // Calculate the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Iterate through elements with a stride pattern to cover large data sizes
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        // Efficiently clamp the input value between min_val and max_val
        float val = in[i];
        if (val < min_val) {
            out[i] = min_val;
        } else if (val > max_val) {
            out[i] = max_val;
        } else {
            out[i] = val;
        }
    }
}