#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adagrad_update_1D_1D(float* x, float* d, float* m, float clip, float lr, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use shared memory to minimize global memory accesses
    __shared__ float shared_d[256];
    __shared__ float shared_m[256];
    
    for (; tid < size; tid += stride) {
        // Load data into shared memory
        shared_d[threadIdx.x] = d[tid];
        shared_m[threadIdx.x] = m[tid];

        __syncthreads();

        // Clip gradient
        if (shared_d[threadIdx.x] > clip) 
            shared_d[threadIdx.x] = clip;
        if (shared_d[threadIdx.x] < -clip) 
            shared_d[threadIdx.x] = -clip;

        // Update m and x using shared memory values
        shared_m[threadIdx.x] += shared_d[threadIdx.x] * shared_d[threadIdx.x];
        x[tid] -= lr * shared_d[threadIdx.x] / sqrt(shared_m[threadIdx.x] + 0.00000001);

        // Reset gradient
        shared_d[threadIdx.x] = 0;

        __syncthreads();

        // Write shared memory data back to global memory
        m[tid] = shared_m[threadIdx.x];
        d[tid] = shared_d[threadIdx.x];
    }
}