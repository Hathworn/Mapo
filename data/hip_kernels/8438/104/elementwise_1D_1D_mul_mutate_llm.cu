#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_mul_mutate(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Eliminate redundant boundary check by restructuring the loop
    for (; tid < size; tid += stride) {
        out[tid] += in1[tid] * in2[tid];
    }
}