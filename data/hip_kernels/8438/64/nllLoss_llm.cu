#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nllLoss(float *x, int x_stride, float *y, int* target) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < gridDim.x * blockDim.x) {  // Boundary check
        int offset = tid * x_stride + target[tid];
        y[tid] = -x[offset];  // Simplified multiplication
    }
}