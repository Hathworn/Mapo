#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void repeat0(float* in, float* out, int outStride0, int outStride1, int outScalarCount) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use while loop to avoid recomputing tid
    while (tid < outScalarCount) {
        int linearIndex = tid;
        int outIndex0 = linearIndex / outStride0;
        linearIndex = linearIndex % outStride0; // Use modulo for efficiency
        int outIndex1 = linearIndex / outStride1;
        int outIndex2 = linearIndex % outStride1; // Use modulo for efficiency
        int inIndex = outIndex2 + (outIndex0 + outIndex1) * outStride1;
        out[tid] = in[inIndex];
        
        // Increment by stride
        tid += stride;
    }
}