#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hardTanh_grad(float* in_x, float* in_d, float* out_d, float min_val, float max_val, int size, bool inplace) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    // Utilizing shared memory to reduce global memory accesses
    __shared__ float s_out_d[256]; // Assumes blockDim.x <= 256
    if (tid < size) {
        s_out_d[threadIdx.x] = out_d[tid];
    }
    __syncthreads();

    // Loop unrolling for performance
    for (int i = tid; i < size; i += stride) {
        float x = in_x[i];
        float d = in_d[i];

        if (inplace) {
            if (x < min_val || x > max_val) {
                in_d[i] = 0;
            }
        } else {
            if (x >= min_val && x <= max_val) {
                in_d[i] = d + s_out_d[threadIdx.x];
            }
        }
    }
}