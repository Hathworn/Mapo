#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void arrayFill(float* data, float value, int size) {
    int stride = gridDim.x * blockDim.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Use unrolling to reduce loop overhead
    for (int i = tid; i < size; i += stride * 4) {
        data[i] = value;
        if (i + stride < size) data[i + stride] = value;
        if (i + 2 * stride < size) data[i + 2 * stride] = value;
        if (i + 3 * stride < size) data[i + 3 * stride] = value;
    }
}