#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void repeat0(float* in, float* out, int outStride0, int outStride1, int outScalarCount) {
    // Optimize kernel by using shared memory and loop unrolling
    extern __shared__ float sharedMem[];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Load data into shared memory
    if (tid < outScalarCount) {
        sharedMem[threadIdx.x] = in[tid];
    }
    __syncthreads();

    for (; tid < outScalarCount; tid += stride) {
        int linearIndex = tid;
        int outIndex0 = linearIndex / outStride0;
        linearIndex -= outIndex0 * outStride0;
        int outIndex1 = linearIndex / outStride1;
        int outIndex2 = linearIndex - outIndex1 * outStride1;
        int inIndex = outIndex2 + (outIndex0 + outIndex1) * outStride1;
        out[tid] = sharedMem[inIndex]; // Using shared memory
    }
}