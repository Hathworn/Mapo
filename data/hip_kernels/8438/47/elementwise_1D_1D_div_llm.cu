#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void elementwise_1D_1D_div(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use grid-stride loop for better memory coalescing
    for (; tid < size; tid += stride) {
        // Ensure index is within array bounds
        if (tid < size) {
            out[tid] = in1[tid] / in2[tid];
        }
    }
}