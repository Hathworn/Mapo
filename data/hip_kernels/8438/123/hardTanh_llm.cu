#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hardTanh(float* in, float* out, float min_val, float max_val, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid stride loop for better utilization
    for (int i = tid; i < size; i += gridDim.x * blockDim.x) {
        // Clamp the input value between min_val and max_val
        out[i] = fminf(fmaxf(in[i], min_val), max_val);
    }
}