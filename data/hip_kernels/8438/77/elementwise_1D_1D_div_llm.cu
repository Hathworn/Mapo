#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_div(float* in1, float* in2, float* out, int size) {
    // Use shared memory if beneficial, though not applicable here due to control divergence 
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Unroll the loop for performance improvement
    for (; tid < size; tid += stride * 4) {
        if (tid < size) out[tid] = in1[tid] / in2[tid];
        if (tid + stride < size) out[tid + stride] = in1[tid + stride] / in2[tid + stride];
        if (tid + 2 * stride < size) out[tid + 2 * stride] = in1[tid + 2 * stride] / in2[tid + 2 * stride];
        if (tid + 3 * stride < size) out[tid + 3 * stride] = in1[tid + 3 * stride] / in2[tid + 3 * stride];
    }
}