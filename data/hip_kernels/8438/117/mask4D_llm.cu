#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask4D(float* in, int* mask, int xstrides0, int xstrides1, int xstrides2, int xstrides3, int scalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use registers for frequently used calculations
    while (tid < scalarCount) {
        int linearIndex = tid;
        int xindex0 = linearIndex / xstrides0;
        linearIndex -= xstrides0 * xindex0;
        int xindex1 = linearIndex / xstrides1;
        linearIndex -= xstrides1 * xindex1;
        int xindex2 = linearIndex / xstrides2;
        int xindex3 = linearIndex - xstrides2 * xindex2;
        
        // Use mask array read coalescing
        if (xindex3 >= __ldg(&mask[xindex0])) 
            in[tid] = 0;

        tid += stride;
    }
}