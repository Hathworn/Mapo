#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nllLoss(float *x, int x_stride, float *y, int* target) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;

    // Calculate the global offset once
    int offset = tid * x_stride + target[tid];

    // Perform operation using memory coalescing
    y[tid] = -x[offset];
}