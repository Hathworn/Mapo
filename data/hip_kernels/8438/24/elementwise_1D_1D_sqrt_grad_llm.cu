#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_sqrt_grad(float* in_x, float* in_d, float* out_x, float* out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unrolling the loop to improve performance
    for (; tid < size; tid += stride) {
        if (tid < size) {
            float temp = out_d[tid] / out_x[tid] / 2; // Reduce redundant computations
            in_d[tid] += temp;
        }

        // Process the next element if within bounds
        if ((tid + stride) < size) {
            float next_tid = tid + stride;
            temp = out_d[next_tid] / out_x[next_tid] / 2; // Reduce redundant computations
            in_d[next_tid] += temp;
        }
    }
}