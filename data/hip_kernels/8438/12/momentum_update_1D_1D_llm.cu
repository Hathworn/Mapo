#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void momentum_update_1D_1D(float* x, float* d, float* m, float learning_rate, float momentum, float gradClip, bool nesterov, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (; tid < size; tid += stride) {
        // Load gradient into register
        float grad = d[tid];

        // Gradient clipping
        grad = fminf(fmaxf(grad, -gradClip), gradClip);

        // Momentum calculation
        float velocity = m[tid] * momentum + grad;

        // Update momentum
        m[tid] = velocity;

        // Nesterov update check
        if (nesterov) {
            // Adjust velocity if using Nesterov momentum
            velocity = grad + momentum * velocity;
        }

        // Update position with learning rate and velocity
        x[tid] -= learning_rate * velocity;

        // Reset gradient
        d[tid] = 0.0f;
    }
}