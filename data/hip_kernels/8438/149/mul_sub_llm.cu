#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_sub(float* in1, float* in2, float* out, int in1ScalarCount, int in2ScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use a while loop for potentially fewer instructions
    while (tid < in1ScalarCount) {
        out[tid] = in1[tid] * in2[tid % in2ScalarCount];
        tid += stride;
    }
}