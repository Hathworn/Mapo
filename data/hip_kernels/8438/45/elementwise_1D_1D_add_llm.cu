#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwise_1D_1D_add(float* in1, float* in2, float* out, int size) {
    // Calculate unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Avoid unnecessary conditional check in loop
    if (tid < size) {
        // Iterating with stride
        int stride = gridDim.x * blockDim.x;
        for (; tid < size; tid += stride) {
            out[tid] = in1[tid] + in2[tid];
        }
    }
}