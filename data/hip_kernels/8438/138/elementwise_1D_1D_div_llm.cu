#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_div(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Thread index for processing
    while (tid < size) {
        // Perform safe division
        out[tid] = in2[tid] != 0.0f ? in1[tid] / in2[tid] : 0.0f;
        tid += stride; // Move to the next element processed by this thread
    }
}