#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_minus(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop unrolling for improved performance
    for (; tid + 3 < size; tid += stride * 4) {
        out[tid] = in1[tid] - in2[tid];
        out[tid + 1] = in1[tid + 1] - in2[tid + 1];
        out[tid + 2] = in1[tid + 2] - in2[tid + 2];
        out[tid + 3] = in1[tid + 3] - in2[tid + 3];
    }

    // Handle remaining elements
    for (; tid < size; tid += stride) {
        out[tid] = in1[tid] - in2[tid];
    }
}