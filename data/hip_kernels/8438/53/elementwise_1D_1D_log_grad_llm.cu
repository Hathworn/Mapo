#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_log_grad(float* in_x, float* in_d, float* out_x, float* out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop unrolling to improve performance by reducing loop overhead
    for (; tid + 4 <= size; tid += stride * 4) {
        in_d[tid] += out_d[tid] / in_x[tid];
        in_d[tid + 1] += out_d[tid + 1] / in_x[tid + 1];
        in_d[tid + 2] += out_d[tid + 2] / in_x[tid + 2];
        in_d[tid + 3] += out_d[tid + 3] / in_x[tid + 3];
    }

    // Handle remaining elements
    for (; tid < size; tid += stride) {
        in_d[tid] += out_d[tid] / in_x[tid];
    }
}