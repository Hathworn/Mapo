#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_mul(float* in1, float* in2, float* out, int size) {
    // Calculate thread ID and stride
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use a loop to cover the entire array
    for (int i = tid; i < size; i += stride) {
        out[i] = in1[i] * in2[i]; // Perform element-wise multiplication
    }
}