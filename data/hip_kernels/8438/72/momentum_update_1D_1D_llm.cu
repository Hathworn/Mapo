#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void momentum_update_1D_1D(float* __restrict__ x, float* __restrict__ d, float* __restrict__ m, float learning_rate, float momentum, float gradClip, bool nesterov, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll loop for better performance
    for (; tid < size; tid += stride) {
        float temp = d[tid];

        // Use ternary operators for better readability
        temp = (temp > gradClip) ? gradClip : ((temp < -gradClip) ? -gradClip : temp);

        m[tid] = m[tid] * momentum + temp;

        // Simplify condition with ternary operator
        x[tid] -= learning_rate * (nesterov ? temp + momentum * m[tid] : m[tid]);

        d[tid] = 0; // Reset d[tid] to zero
    }
}