#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void hardTanh(float* in, float* out, float min_val, float max_val, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Optimize thread index calculation for readability
    int stride = blockDim.x * gridDim.x; // Reorder for consistency
    for (int i = tid; i < size; i += stride) { // Iterate over all elements with stride
        float val = in[i]; // Read input once to register
        out[i] = (val < min_val) ? min_val : (val > max_val ? max_val : val); // Use ternary operator efficiently
    }
}