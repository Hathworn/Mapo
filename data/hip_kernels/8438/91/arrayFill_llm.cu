#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void arrayFill(float* data, float value, int size) {
    // Use a shared array to enhance memory access
    __shared__ float sharedValue;
    if (threadIdx.x == 0) sharedValue = value; // Initialize shared memory with value
    __syncthreads(); // Sync threads to ensure sharedValue is set

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    // Improved loop access with shared memory
    for (int i = tid; i < size; i += stride) {
        data[i] = sharedValue; // Use shared memory value
    }
}