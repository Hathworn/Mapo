#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hardTanh_grad(float* in_x, float* in_d, float* out_d, float min_val, float max_val, int size, bool inplace) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = tid; i < size; i += stride) {
        float x_val = in_x[i];
        // Avoid redundant memory accesses
        if (inplace) {
            in_d[i] = (x_val < min_val || x_val > max_val) ? 0 : in_d[i];
        } else {
            if (x_val >= min_val && x_val <= max_val) {
                in_d[i] += out_d[i];
            }
        }
    }
}