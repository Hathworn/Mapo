#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_mul(float* in1, float* in2, float* out, int size) {
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure within bounds before processing
    if (tid < size) {
        out[tid] = in1[tid] * in2[tid];
    }
}