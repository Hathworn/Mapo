#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask4D(float* in, int* mask, int xstrides0, int xstrides1, int xstrides2, int xstrides3, int scalarCount) {
    // Calculate the thread's unique identifier
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
  
    // Loop over data with thread-stride
    for (; tid < scalarCount; tid += stride) {
        int linearIndex = tid;

        // Unroll index calculation to reduce repetitive division
        int xindex0 = linearIndex / xstrides0;
        linearIndex -= xstrides0 * xindex0;
        int xindex1 = __fdiv_rd(linearIndex, xstrides1);  // Use fast integer division
        linearIndex -= xstrides1 * xindex1;
        int xindex2 = __fdiv_rd(linearIndex, xstrides2);
        int xindex3 = linearIndex - xstrides2 * xindex2;

        // Apply mask condition
        if (xindex3 >= mask[xindex0]) in[tid] = 0;
    }
}