#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nllLoss(float *x, int x_stride, float *y, int* target) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread ID is within bounds
    if (tid < gridDim.x * blockDim.x) {
        int offset = tid * x_stride + target[tid];
        y[tid] = -x[offset]; // Simplified negative multiplication
    }
}