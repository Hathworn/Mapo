#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_div(float* in1, float* in2, float* out, int size) {
    // calculate the thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // loop through elements using the stride
    for (; tid < size; tid += stride) {
        // perform division if index is within bounds
        float val2 = in2[tid];
        // add a check to avoid division by zero
        out[tid] = (val2 != 0.0f) ? (in1[tid] / val2) : 0.0f;
    }
}