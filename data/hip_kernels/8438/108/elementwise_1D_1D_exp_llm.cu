#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_exp(float* in, float* out, int size) {
    // Compute the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use blockDim.x for loop stride, improves memory access efficiency
    int stride = blockDim.x;

    // Changed loop condition to prevent unnecessary checks
    for (int i = tid; i < size; i += stride * gridDim.x) {
        out[i] = exp(in[i]);
    }
}