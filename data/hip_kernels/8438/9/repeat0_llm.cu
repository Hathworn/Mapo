#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void repeat0(float* in, float* out, int outStride0, int outStride1, int outScalarCount) {
    // Improve performance by using the total thread count once
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    while (tid < outScalarCount) {
        int linearIndex = tid;
        int outIndex0 = linearIndex / outStride0;
        linearIndex = linearIndex - outIndex0 * outStride0;
        int outIndex1 = linearIndex / outStride1;
        int outIndex2 = linearIndex - outIndex1 * outStride1;
        // Calculate the input index efficiently
        int inIndex = outIndex2 + (outIndex0 + outIndex1) * outStride1;
        out[tid] = in[inIndex];
        // Increment this specific thread's index directly
        tid += stride;
    }
}