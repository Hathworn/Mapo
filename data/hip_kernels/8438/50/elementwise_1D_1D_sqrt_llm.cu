#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_sqrt(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Unrolling loop to improve performance
    for (; tid + 3 < size; tid += stride * 4) {
        out[tid] = sqrt(in[tid]);
        out[tid + 1] = sqrt(in[tid + 1]);
        out[tid + 2] = sqrt(in[tid + 2]);
        out[tid + 3] = sqrt(in[tid + 3]);
    }
    // Handle remaining elements
    for (; tid < size; tid += stride) {
        out[tid] = sqrt(in[tid]);
    }
}