#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask4D(float* in, int* mask, int xstrides0, int xstrides1, int xstrides2, int xstrides3, int scalarCount) {
    // Calculate thread ID and stride efficiently using grid-stride loop
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop through elements, updating by stride
    for (int idx = tid; idx < scalarCount; idx += stride) {
        int linearIndex = idx;
        // Calculate multi-dimensional indices using modulo and division
        int xindex0 = linearIndex / xstrides0;
        linearIndex -= xindex0 * xstrides0;
        int xindex1 = linearIndex / xstrides1;
        linearIndex -= xindex1 * xstrides1;
        int xindex2 = linearIndex / xstrides2;
        int xindex3 = linearIndex % xstrides2; // Use modulo for xindex3 calculation

        // Apply mask condition
        if (xindex3 >= mask[xindex0]) {
            in[idx] = 0;
        }
    }
}