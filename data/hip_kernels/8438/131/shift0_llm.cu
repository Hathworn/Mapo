#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shift0(float* in, float* out, int inDim0, int inStride0, int inStride1, int inScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (; tid < inScalarCount; tid += stride) {
        int inIndex0 = tid / inStride0;
        int inIndex1 = (tid % inStride0) / inStride1; // Use modulus for better readability and efficiency
        if (inIndex0 + inIndex1 >= inDim0) break; // Use break instead of return for better flow control
        out[tid + inIndex1 * inStride0] = in[tid];
    }
}