#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_sub_grad(float* in1_x, float* in1_d, float* in2_x, float* in2_d, float* out, int in1ScalarCount, int in2ScalarCount) {
    // Calculate unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Process multiple elements per thread
    while (tid < in1ScalarCount) {
        int index = tid % in2ScalarCount;
        
        // Perform operations with reduced memory accesses
        float out_val = out[tid];
        in1_d[tid] += out_val * in2_x[index];
        in2_d[tid] = in1_x[tid] * out_val;  // Utilize registers for repeated values

        // Move to the next element this thread should process
        tid += stride;
    }
}