#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_log_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop unrolling for better performance
    for (; tid < size; tid += stride * 4) {
        if (tid < size) in_d[tid] += out_d[tid] / in_x[tid];
        if (tid + stride < size) in_d[tid + stride] += out_d[tid + stride] / in_x[tid + stride];
        if (tid + 2 * stride < size) in_d[tid + 2 * stride] += out_d[tid + 2 * stride] / in_x[tid + 2 * stride];
        if (tid + 3 * stride < size) in_d[tid + 3 * stride] += out_d[tid + 3 * stride] / in_x[tid + 3 * stride];
    }
}