#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_minus(float* in1, float* in2, float* out, int size) {
    // Calculate global index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is within bounds
    if (tid < size) {
        // Perform element-wise subtraction
        out[tid] = in1[tid] - in2[tid];
    }
}