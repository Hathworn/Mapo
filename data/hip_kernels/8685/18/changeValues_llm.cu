#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void changeValues(float *matrix, int size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Check if index is within bounds and handle warp divergence
    if (index < size) {
        float value = matrix[index] * 10;

        // Directly cast without using extra variable 'b'
        matrix[index] = static_cast<float>(static_cast<int>(value));
    }
}