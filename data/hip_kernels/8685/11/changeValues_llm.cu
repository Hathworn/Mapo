#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void changeValues(float *matrix, int size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Check within bounds before accessing the matrix
    if (index < size) {
        // Directly cast the scaled value to int and back to float
        matrix[index] = (float)((int)(matrix[index] * 10));
    }
}