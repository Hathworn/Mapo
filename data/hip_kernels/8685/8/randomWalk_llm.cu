#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomWalk(double *results, int T, int N) {
    // Use thread ID to initialize state for parallel execution
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprandState_t state;
    hiprand_init (1234, idx, 0, &state); // Unique state for each thread

    // Ensure results initialized properly
    results[idx * N] = 0.0;

    double random;
    for (int j = 1; j < N; j++) {
        random = hiprand_normal_double(&state);
        // Remove computation from index calculation for efficiency
        double delta = random * sqrt((double) T / N);
        results[idx * N + j] = results[idx * N + j - 1] + delta;
    }

    /*
    Generate 2 doubles at once. Test later to see if this is more efficient:
    double hiprand_normal2_double (state);
    */
}
```
