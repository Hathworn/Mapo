#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void changeValues(float *matrix, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        // Perform multiplication and type conversion in one step
        matrix[index] = float(int(matrix[index] * 10));
    }
}