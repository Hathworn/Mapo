#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void changeValues(double *matrix, int size) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (index < size) {
        // Use fma for better performance and avoid unnecessary type casting
        matrix[index] = trunc(matrix[index] * 10.0);
    }
}