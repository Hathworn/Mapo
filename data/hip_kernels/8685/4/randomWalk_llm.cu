#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomWalk(double *results, int *crossTimes, int T, int N, int numSims) {
    int crossTimeIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (crossTimeIndex < numSims) {
        // Initialize HIP random state using unique index
        hiprandState_t state;
        hiprand_init(1234, crossTimeIndex, 0, &state);

        double random;
        int start = crossTimeIndex * N;

        crossTimes[crossTimeIndex] = 0;
        results[start] = 0.0;

        // Optimize: Unroll loop for better performance
        #pragma unroll 2
        for (int j = start + 1; j < start + N; j++) {
            random = hiprand_normal_double(&state);
            results[j] = results[j-1] + random * sqrt((double) T / N);
        }
    }

    /*
    Generate 2 doubles at once. Test later to see if this is more efficient:
    double hiprand_normal2_double (state);
    */
}