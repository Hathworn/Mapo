#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void changeValues(float *matrix, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        // Use native HIP function for faster computation
        matrix[index] = floorf(matrix[index] * 10);
    }
}