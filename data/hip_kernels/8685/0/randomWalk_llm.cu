#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

int *crossTimes = nullptr;

struct DataStruct {
    int deviceID;
    int sims;
    double *resultArray;
    int *crossArray;
    int N;
    double T;
    double drift;
    double lowerThreshold;
    double upperThreshold;
};

__global__ void randomWalk(double *results, int *crossTimes, int T, int N, double drift, int numSims, double lowerThreshold, double upperThreshold, double deviceID) {
    int crossTimeIndex = threadIdx.x + blockIdx.x * blockDim.x;

    if (crossTimeIndex < numSims) {
        // Initialize random number generator with unique seed
        hiprandState_t state;
        hiprand_init(blockIdx.x * (1000 * deviceID) + threadIdx.x + clock64(), 0, 0, &state);
        double random;
        
        int start = crossTimeIndex * N;  // Simplified start index calculation

        crossTimes[crossTimeIndex] = 0;
        results[start] = 0.0;
        bool crossed = false;

        // Calculate constants outside the loop
        double sqrt_T_div_N = sqrt((double)T / N);
        double drift_T_div_N = drift * ((double)T / N);

        for (int j = 1; j < N; j++) {
            random = hiprand_normal_double(&state);
            results[start + j] = results[start + j - 1] + drift_T_div_N + random * sqrt_T_div_N;

            if (!crossed) {
                if (results[start + j] >= upperThreshold) {
                    crossTimes[crossTimeIndex] = j;
                    crossed = true;
                } else if (results[start + j] <= lowerThreshold) {
                    crossTimes[crossTimeIndex] = -j;
                    crossed = true;
                }
            }
        }
    }
}