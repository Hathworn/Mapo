#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void squeeze(float *B, int dim, int length, float L, float M) {
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the threadId is within required bounds
    if (threadId < dim) {
        int index = threadId + length; // Calculate the correct index position

        // Perform calculation for the given index
        B[index] = 1 / (1 + expf(-L * (B[index] - M)));
    }
}