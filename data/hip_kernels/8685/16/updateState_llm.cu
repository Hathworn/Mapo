#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateState(double *B, double *external, double *lamBeta, int dim, float timestep, double noise, int length, int totalIterations, int iterationNum) {
    int index = (blockIdx.x * blockDim.x) + threadIdx.x + length;
    if (index >= length && index < length + dim) {
        int neuronNum = index % dim;
        double lam = lamBeta[neuronNum * 2];
        double beta = lamBeta[neuronNum * 2 + 1];

        double input = B[index] + external[neuronNum * (totalIterations) + iterationNum];
        double old_output = B[index - dim];
        double d_layers = (-1 * old_output) + 1 / (1 + expf(-1 * lam * (input - beta)));

        // Improve random number generation by initializing state outside loops
        __shared__ hiprandState_t sharedState[256]; // assuming max of 256 threads/block
        if (threadIdx.x < 256) {
            hiprand_init(blockIdx.x * 1000 + threadIdx.x, 0, 0, &sharedState[threadIdx.x]);
        }
        __syncthreads();

        hiprandState_t state = sharedState[threadIdx.x];
        float random = hiprand_normal(&state);
        sharedState[threadIdx.x] = state;
        double guassian_noise = noise * random * sqrt(timestep);
        
        B[index] = old_output + d_layers * timestep + guassian_noise;
    }
}