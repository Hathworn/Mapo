#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomWalk(double *results, int *crossTimes, int T, int N, int numSims, double lowerThreshold, double upperThreshold, int *dev_failCross, double seconds) {
    int crossTimeIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if (crossTimeIndex >= numSims) return;  // Early exit for out of bounds
    
    // Initialize random state
    hiprandState_t state;
    hiprand_init(blockIdx.x * 1000 + threadIdx.x + static_cast<int>(seconds), 0, 0, &state);
    
    int start = crossTimeIndex * N;
    crossTimes[crossTimeIndex] = 0;
    results[start] = 0.0;
    bool crossed = false;
    double random;

    for (int j = 1; j < N; j++) {  // Start loop from 1 to reduce index calculations
        random = hiprand_normal_double(&state);
        results[start + j] = results[start + j - 1] + random * sqrt(static_cast<double>(T) / N);
        
        if (!crossed && results[start + j] >= upperThreshold) {
            crossTimes[crossTimeIndex] = j;
            crossed = true;
        } else if (!crossed && results[start + j] <= lowerThreshold) {
            crossTimes[crossTimeIndex] = -j;
            crossed = true;
        }
    }

    // If no crossing occurred
    if (!crossed) {
        atomicAdd(dev_failCross, 1);
    }
}