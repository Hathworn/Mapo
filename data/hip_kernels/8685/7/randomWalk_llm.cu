#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomWalk(double *results, int T, int N) {
    extern __shared__ double sharedResults[]; // Use shared memory for results
    int threadId = threadIdx.x;
    int blockSize = blockDim.x;
    int i, start = threadId;

    hiprandState_t state;
    hiprand_init(1234, threadId, 0, &state); // Use thread-specific state
    double random;

    if (start == 0) {
        sharedResults[0] = 0.0;
    }
    __syncthreads(); // Ensure initial value is set

    for (i = start + 1; i < N; i += blockSize) {
        random = hiprand_normal_double(&state);
        if (i < N) { // Ensure not out of bounds
            sharedResults[i] = (i == 1) ? random * sqrt((double) T / N) 
                                        : sharedResults[i - 1] + random * sqrt((double) T / N);
        }
    }

    __syncthreads(); // Allow all threads to complete writing to shared memory

    if (start < N) {
        results[start] = sharedResults[start]; // Copy from shared memory to global memory
    }
}