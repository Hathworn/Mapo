#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateState(float *B, float *external, int dim, float timestep, int length, float L, float M) {
    int index = blockIdx.x * blockDim.x + threadIdx.x + length;

    // Ensure index is within bounds for computation
    if (index < length + dim) {
        float input = B[index] + external[index];
        float old_output = B[index - dim];

        // Optimize the calculation by combining operations for better performance
        float exp_val = expf(-L * (input - M));
        float d_layers = (-old_output) + 1 / (1 + exp_val);

        // Update B using optimized calculation
        B[index] = old_output + d_layers * timestep;
    }
}