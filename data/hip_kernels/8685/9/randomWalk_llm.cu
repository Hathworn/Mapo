#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomWalk(double *results, int *crossTimes, int T, int N, int numSims, double upperThreshold, double deviceID) {
    // Calculate linear thread index.
    int crossTimeIndex = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure thread is within bounds.
    if (crossTimeIndex < numSims) {
        // Initialize RNG state with better randomness using deviceID as a seed.
        hiprandState_t state;
        hiprand_init(blockIdx.x * (1000 * deviceID) + threadIdx.x + clock64(), 0, 0, &state);

        // Determine start index for results array.
        int start = crossTimeIndex * N;

        // Initialize crossing times and results.
        crossTimes[crossTimeIndex] = 0;
        results[start] = 0.0;

        bool crossed = false; // Indicates if threshold has been crossed.

        // Pre-calculate the factor for random walk.
        double stepFactor = sqrt((double)T / N);

        for (int j = 1; j < N; j++) {
            // Generate random step and update path.
            double random = hiprand_normal_double(&state);
            results[start + j] = results[start + j - 1] + random * stepFactor;

            // Check for crossing the upper threshold.
            if (!crossed && results[start + j] >= upperThreshold) {
                crossTimes[crossTimeIndex] = j; // Record crossing time.
                crossed = true;
            }
        }
    }
}