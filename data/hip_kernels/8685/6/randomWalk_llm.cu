#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void randomWalk(double *results, int *crossTimes, double T, int N, int numSims, double lowerThreshold, double upperThreshold, int *dev_failCross, double seconds) {
    int crossTimeIndex = threadIdx.x + blockIdx.x * blockDim.x;

    if (crossTimeIndex < numSims) {
        // Use static shared memory to eliminate redundant initialization
        __shared__ hiprandState_t shared_state;
        
        // Initialize the random state in shared memory
        if (threadIdx.x == 0) {
            hiprand_init(blockIdx.x * 1000 + seconds, 0, 0, &shared_state);
        }
        __syncthreads();

        hiprandState_t localState = shared_state; // All threads copy static shared state to local
        double random;
        int start = crossTimeIndex * N;
        bool crossed = false;

        crossTimes[crossTimeIndex] = 0;
        results[start] = 0.0;

        // Loop unrolling for improved performance
        #pragma unroll
        for (int j = start + 1; j < start + N; j++) {
            random = hiprand_normal_double(&localState);
            results[j] = results[j-1] + random * sqrt((double) T / N);
            if (results[j] >= upperThreshold && !crossed) {
                crossTimes[crossTimeIndex] = j - start;
                crossed = true;
            } else if (results[j] <= lowerThreshold && !crossed) {
                crossTimes[crossTimeIndex] = -1 * (j - start);
                crossed = true;
            }
        }

        if (!crossed) {
            atomicAdd(dev_failCross, 1);
        }
    }
}