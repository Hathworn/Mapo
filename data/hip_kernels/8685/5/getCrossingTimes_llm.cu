#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getCrossingTimes(double *results, int *crossTimes, int N, int numSims, int lowerThreshold, int upperThreshold) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Optimize the loop increment for better memory access pattern
    int totalThreads = blockDim.x * gridDim.x;

    while (tid < N * numSims) {
        int simIndex = tid / N;
        if (crossTimes[simIndex] == 0) {
            double resultVal = results[tid]; // Avoid redundant memory accesses
            if (resultVal <= lowerThreshold || resultVal >= upperThreshold) {
                crossTimes[simIndex] = tid % N;
            }
        }
        tid += totalThreads; // Use totalThreads for better workload distribution
    }
}