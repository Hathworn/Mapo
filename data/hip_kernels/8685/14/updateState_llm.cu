#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateState(float *B, float *external, int dim, float timestep, float noise, int length, int totalIterations, int iterationNum, float L, float M) {
    // Calculate global thread index
    int index = (blockIdx.x * blockDim.x) + threadIdx.x + length;

    // Ensure index is within valid range
    if (index >= length && index < length + dim) {
        // Calculate neuron number based on index
        int neuronNum = index % dim;
        
        // Calculate neuron input and old output
        float input = B[index] + external[neuronNum * totalIterations + iterationNum];
        float old_output = B[index - dim];
        
        // Calculate differential layers using sigmoid function
        float d_layers = -old_output + 1 / (1 + expf(-L * (input - M)));

        // Use thread-specific state for random number generation        
        __shared__ hiprandState_t sharedState[blockDim.x];
        if (threadIdx.x == 0) {
            hiprand_init(blockIdx.x * 1000 + clock64(), 0, 0, &sharedState[threadIdx.x]);
        }
        __syncthreads();

        // Generate Gaussian noise with pre-seeded random state
        hiprandState_t localState = sharedState[threadIdx.x];
        float random = hiprand_normal(&localState);
        float gaussian_noise = noise * random * sqrt(timestep);

        // Update neuron state
        B[index] = old_output + d_layers * timestep + gaussian_noise;

        // Write back updated random state to shared memory
        sharedState[threadIdx.x] = localState;
    }
}