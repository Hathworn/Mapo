#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void squeeze(float *B, int dim, int length, float L, float M) {
    // Calculate the global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x + length;
    
    // Use only valid threads
    if (index < length + dim) {
        // Pre-calculate the displacement to reduce redundant computation
        float temp_val = -(L * (B[index] - M));
        B[index] = 1.0f / (1.0f + expf(temp_val));
    }
}