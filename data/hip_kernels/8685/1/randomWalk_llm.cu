#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#include <hiprand_kernel.h>

__global__ void randomWalk(double *results, int *crossTimes, int T, int N, double drift, int numSims, double lowerThreshold, double upperThreshold, int deviceID) {

    int crossTimeIndex = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure thread is within bounds
    if (crossTimeIndex < numSims) {
        
        // Use shared memory for better performance
        __shared__ hiprandState_t state[1024];
        
        // Initialize random state for each thread using thread-specific seeds
        hiprand_init(clock64() + crossTimeIndex * (1000 * deviceID), 0, 0, &state[threadIdx.x]);
        
        double random;
        int start = crossTimeIndex * N; // Use crossTimeIndex directly for start calculation
        
        crossTimes[crossTimeIndex] = 0;
        results[start] = 0.0;
        bool crossed = false;

        // Precompute factors outside the loop
        double sqrtFactor = sqrt((double)T / N);
        double driftFactor = drift * ((double)T / N);

        for (int j = 1; j < N; j++) { // Simplified loop by using j directly
            random = hiprand_normal_double(&state[threadIdx.x]);
            results[start + j] = results[start + j - 1] + driftFactor + random * sqrtFactor;
            if (!crossed) {
                if (results[start + j] >= upperThreshold) {
                    crossTimes[crossTimeIndex] = j;
                    crossed = true;
                } else if (results[start + j] <= lowerThreshold) {
                    crossTimes[crossTimeIndex] = -j;
                    crossed = true;
                }
            }
        }
    }
}