#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void randomWalk(double *results, int *crossTimes, int T, int N, int numSims, double upperThreshold, double deviceID) {

    // Calculate the global index
    int crossTimeIndex = threadIdx.x + blockIdx.x * blockDim.x;

    if (crossTimeIndex < numSims) {

        // Initialize the random number generator using a unique seed
        hiprandState_t state;
        hiprand_init(blockIdx.x * (1000 * deviceID) + threadIdx.x + clock64(), 0, 0, &state);
        
        // Starting position of this simulation in results array
        int start = crossTimeIndex * N;

        // Set default value of cross time for this simulation to 0
        crossTimes[crossTimeIndex] = 0;

        // Starting point of path is 0
        results[start] = 0.0;

        // Track whether this path has crossed the threshold
        bool crossed = false;

        // Calculate next step of path, break early if crossed
        for (int j = 1; j < N; j++) {
            double random = hiprand_normal_double(&state);
            results[start + j] = results[start + j - 1] + random * sqrt((double) T / N);

            if (!crossed && results[start + j] >= upperThreshold) {
                crossTimes[crossTimeIndex] = j;
                crossed = true;
                break; // Exit loop early if threshold crossed
            }
        }
    }
}
