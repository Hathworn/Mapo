#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void one_channel_mul_kernel(float *data_l, float *data_r, float *result)
{
    // Compute linear thread index for improved performance
    int threadId = 2 * (blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x);
    int one_ch_index = 2 * (threadIdx.y * blockDim.x + threadIdx.x);

    // Use shared memory for faster data access
    extern __shared__ float shared_data_r[];
    shared_data_r[one_ch_index] = data_r[one_ch_index];
    shared_data_r[one_ch_index + 1] = data_r[one_ch_index + 1];
    __syncthreads();

    // Perform operations using shared memory
    result[threadId] = data_l[threadId] * shared_data_r[one_ch_index] - data_l[threadId + 1] * shared_data_r[one_ch_index + 1];
    result[threadId + 1] = data_l[threadId] * shared_data_r[one_ch_index + 1] + data_l[threadId + 1] * shared_data_r[one_ch_index];
}