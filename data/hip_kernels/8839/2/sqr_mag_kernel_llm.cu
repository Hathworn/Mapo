#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sqr_mag_kernel(float *data, float *result) 
{
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;  // Use single dimension for block and thread indexing
    int threadId = 2 * globalId; // Calculate actual position in data

    float real = data[threadId];               // Load real part
    float imag = data[threadId + 1];           // Load imaginary part
    result[threadId] = real * real + imag * imag; // Compute squared magnitude
    result[threadId + 1] = 0;                  // Set imaginary part of the result to zero
}