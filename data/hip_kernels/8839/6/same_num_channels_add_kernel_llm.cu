#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void same_num_channels_add_kernel(float *data_l, float *data_r, float *result)
{
    // Calculate unique thread index in flattened grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    int stride = blockDim.x * gridDim.x; // Calculate the total stride
    int index = 2 * idx; // Each index handles two elements

    // Unroll loop to optimize memory access and computation
    for (; index < gridDim.x * gridDim.y * blockDim.x * blockDim.y * 2; index += 2 * stride) {
        result[index] = data_l[index] + data_r[index];
        result[index + 1] = data_l[index + 1] + data_r[index + 1];
    }
}