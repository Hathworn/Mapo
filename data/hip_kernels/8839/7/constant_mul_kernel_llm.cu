#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constant_mul_kernel(float *data_l, float constant, float *result)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index

    // Check bounds before accessing
    if (index * 2 < gridDim.x * blockDim.x * 2) {
        result[index * 2] = data_l[index * 2] * constant;
        result[index * 2 + 1] = data_l[index * 2 + 1] * constant;
    }
}