#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constant_add_kernel(float *data_l, float constant, float *result)
{
    // Optimize: Use linear thread index to improve memory access pattern
    int threadIdxGlobal = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop to enhance performance
    int index1 = threadIdxGlobal * 2;
    int index2 = index1 + 1;

    result[index1] = data_l[index1] + constant;
    result[index2] = data_l[index2];
}