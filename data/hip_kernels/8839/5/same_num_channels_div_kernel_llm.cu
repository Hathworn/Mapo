#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void same_num_channels_div_kernel(float *data_l, float *data_r, float *result)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = 2 * (blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x);
   
    // Precompute repeated expressions to reduce redundant calculations
    float real_r1 = data_r[threadId];
    float real_r2 = data_r[threadId + 1];
    float denom = real_r1 * real_r1 + real_r2 * real_r2;

    // Ensure denominator is non-zero before division
    if (denom != 0) {
        result[threadId] = (data_l[threadId] * real_r1 + data_l[threadId + 1] * real_r2) / denom;
        result[threadId + 1] = (data_l[threadId + 1] * real_r1 - data_l[threadId] * real_r2) / denom;
    } else {
        result[threadId] = 0.0f;
        result[threadId + 1] = 0.0f;
    }
}