#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scales_channel_mul_kernel(float *data_l, float *data_r, float *result)
{
    // Calculate index based on block and thread ID, optimizing index calculation
    int globalThreadId = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    int threadId = 2 * globalThreadId;
    int one_ch_index = 2 * threadIdx.x;

    float left_real = data_l[threadId];
    float left_imag = data_l[threadId + 1];
    float right_real = data_r[one_ch_index];
    float right_imag = data_r[one_ch_index + 1];

    // Perform complex multiplication
    result[threadId] = left_real * right_real - left_imag * right_imag;
    result[threadId + 1] = left_real * right_imag + left_imag * right_real;
}