#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conj_kernel(float *data, float *result)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Optimize calculation of linear index
    int offset = blockIdx.y * gridDim.x * blockDim.x; // Calculate offset for 2D grid
    int threadId = 2 * (offset + idx); // Combine into single index calculation for clarity

    result[threadId] = data[threadId];
    result[threadId + 1] = -data[threadId + 1];
}