#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sqr_norm_kernel(int n, float *out, float *data, float rows, float cols)
{
    extern __shared__ float sdata[];
    // Calculate global thread index
    int globalThreadId = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    int threadId = 2 * globalThreadId;

    int i = threadIdx.y * blockDim.x + threadIdx.x;
    
    // Initialize shared memory data
    sdata[i] = 0;
    
    // Perform calculations
    if(threadId + 1 < n) { // Ensure within bounds
        sdata[i] = data[threadId] * data[threadId] + data[threadId + 1] * data[threadId + 1];
    }
    __syncthreads();

    // Reduce within block
    for (unsigned int s = blockDim.x * blockDim.y / 2; s > 0; s >>= 1) {
        if (i < s) {
            sdata[i] += sdata[i + s];
        }
        __syncthreads();
    }

    // Store result of reduction
    if (i == 0) {
        atomicAdd(&out[blockIdx.x], sdata[0] / (rows * cols));
    }
}