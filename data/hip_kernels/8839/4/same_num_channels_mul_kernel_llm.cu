#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void same_num_channels_mul_kernel(float *data_l, float *data_r, float *result)
{
    // Use linear thread indexing for better coalescing
    int threadId = 2 * (blockDim.x * blockDim.y * blockIdx.y * gridDim.x + blockDim.x * blockIdx.x + blockDim.x * threadIdx.y + threadIdx.x);

    // Reuse intermediate results to reduce redundant calculations
    float a = data_l[threadId];
    float b = data_l[threadId + 1];
    float c = data_r[threadId];
    float d = data_r[threadId + 1];

    result[threadId] = a * c - b * d;
    result[threadId + 1] = a * d + b * c;
}