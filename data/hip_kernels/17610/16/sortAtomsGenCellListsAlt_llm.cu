#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sortAtomsGenCellListsAlt(unsigned int natoms, const float4 *xyzr_d, const float4 *color_d, const unsigned int *atomIndex_d, unsigned int *sorted_atomIndex_d, const unsigned int *atomHash_d, float4 *sorted_xyzr_d, float4 *sorted_color_d, uint2 *cellStartEnd_d) {
    extern __shared__ unsigned int hash_s[]; // blockSize + 1 elements
    unsigned int index = (blockIdx.x * blockDim.x) + threadIdx.x;
    unsigned int hash;

    // Load the hash and initialize shared memory
    if (index < natoms) {
        hash = atomHash_d[index];
        hash_s[threadIdx.x+1] = hash;
        if (index > 0 && threadIdx.x == 0) {
            hash_s[0] = atomHash_d[index-1];
        }
    }

    __syncthreads();

    if (index < natoms) {
        // Check if it's the start of a new cell and set boundaries
        if (index == 0 || hash != hash_s[threadIdx.x]) {
            cellStartEnd_d[hash].x = index;
            if (index > 0)
                cellStartEnd_d[hash_s[threadIdx.x]].y = index;
        }

        if (index == natoms - 1) {
            cellStartEnd_d[hash].y = index + 1;
        }

        // Reorder atoms according to sorted indices
        unsigned int sortedIndex = atomIndex_d[index];
        sorted_atomIndex_d[sortedIndex] = index;
        sorted_xyzr_d[index] = xyzr_d[sortedIndex];

        // Reorder colors according to sorted indices, if provided
        if (color_d != NULL) {
            sorted_color_d[index] = color_d[sortedIndex];
        }
    }
}