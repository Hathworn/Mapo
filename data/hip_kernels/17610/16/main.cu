#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "sortAtomsGenCellListsAlt.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    unsigned int natoms = 1;
const float4 *xyzr_d = NULL;
hipMalloc(&xyzr_d, XSIZE*YSIZE);
const float4 *color_d = NULL;
hipMalloc(&color_d, XSIZE*YSIZE);
const unsigned int *atomIndex_d = NULL;
hipMalloc(&atomIndex_d, XSIZE*YSIZE);
unsigned int *sorted_atomIndex_d = NULL;
hipMalloc(&sorted_atomIndex_d, XSIZE*YSIZE);
const unsigned int *atomHash_d = NULL;
hipMalloc(&atomHash_d, XSIZE*YSIZE);
float4 *sorted_xyzr_d = NULL;
hipMalloc(&sorted_xyzr_d, XSIZE*YSIZE);
float4 *sorted_color_d = NULL;
hipMalloc(&sorted_color_d, XSIZE*YSIZE);
uint2 *cellStartEnd_d = NULL;
hipMalloc(&cellStartEnd_d, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
sortAtomsGenCellListsAlt<<<gridBlock, threadBlock>>>(natoms,xyzr_d,color_d,atomIndex_d,sorted_atomIndex_d,atomHash_d,sorted_xyzr_d,sorted_color_d,cellStartEnd_d);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
sortAtomsGenCellListsAlt<<<gridBlock, threadBlock>>>(natoms,xyzr_d,color_d,atomIndex_d,sorted_atomIndex_d,atomHash_d,sorted_xyzr_d,sorted_color_d,cellStartEnd_d);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
sortAtomsGenCellListsAlt<<<gridBlock, threadBlock>>>(natoms,xyzr_d,color_d,atomIndex_d,sorted_atomIndex_d,atomHash_d,sorted_xyzr_d,sorted_color_d,cellStartEnd_d);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}