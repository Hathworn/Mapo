#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void gaussdensity_direct_alt(int natoms, const float4 *xyzr, float gridspacing, unsigned int z, float *densitygrid) {
    unsigned int xindex = (blockIdx.x * blockDim.x) * DUNROLLX + threadIdx.x;
    unsigned int yindex = (blockIdx.y * blockDim.y) + threadIdx.y;
    unsigned int zindex = (blockIdx.z * blockDim.z) + threadIdx.z;
    unsigned int outaddr =
            ((gridDim.x * blockDim.x) * DUNROLLX) * (gridDim.y * blockDim.y) * zindex +
            ((gridDim.x * blockDim.x) * DUNROLLX) * yindex + xindex;
    zindex += z;

    float coorx = gridspacing * xindex;
    float coory = gridspacing * yindex;
    float coorz = gridspacing * zindex;

    // Initialize density values using unroll factor
    float densityval[DUNROLLX] = {0};

    float gridspacing_coalesce = gridspacing * DBLOCKSZX;

    for (int atomid = 0; atomid < natoms; atomid++) {
        float4 atom = xyzr[atomid];
        float dy = coory - atom.y;
        float dz = coorz - atom.z;
        float dyz2 = dy * dy + dz * dz;

        // Loop to calculate density values for each unroll position
        for (int i = 0; i < DUNROLLX; ++i) {
            float dx = coorx + i * gridspacing_coalesce - atom.x;
            float r2 = (dx * dx + dyz2) * atom.w;
            densityval[i] += exp2f(-r2);
        }
    }

    // Write results to global memory
    for (int i = 0; i < DUNROLLX; ++i) {
        densitygrid[outaddr + i * DBLOCKSZX] += densityval[i];
    }
}