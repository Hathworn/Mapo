#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void gaussdensity_direct_tex(int natoms, const float4 *xyzr, const float4 *colors, float gridspacing, unsigned int z, float *densitygrid, float3 *voltexmap, float invisovalue) {
    unsigned int xindex = (blockIdx.x * blockDim.x) * DUNROLLX + threadIdx.x;
    unsigned int yindex = (blockIdx.y * blockDim.y) + threadIdx.y;
    unsigned int zindex = (blockIdx.z * blockDim.z) + threadIdx.z;
    unsigned int outaddr =
        ((gridDim.x * blockDim.x) * DUNROLLX) * (gridDim.y * blockDim.y) * zindex +
        ((gridDim.x * blockDim.x) * DUNROLLX) * yindex + xindex;
    zindex += z;

    float coorx = gridspacing * xindex;
    float coory = gridspacing * yindex;
    float coorz = gridspacing * zindex;

    float densityvalx[DUNROLLX] = {0}; // Use array to eliminate redundant variables
    float3 densitycolx[DUNROLLX];
    for (int i = 0; i < DUNROLLX; i++) {
        densitycolx[i] = make_float3(0.0f, 0.0f, 0.0f);
    }

    float gridspacing_coalesce = gridspacing * DBLOCKSZX;

    int atomid;
    for (atomid = 0; atomid < natoms; atomid++) {
        float4 atom = xyzr[atomid];
        float4 color = colors[atomid];

        float dy = coory - atom.y;
        float dz = coorz - atom.z;
        float dyz2 = dy * dy + dz * dz;

        float dx[DUNROLLX];
        dx[0] = coorx - atom.x;
        for (int i = 1; i < DUNROLLX; i++) {
            dx[i] = dx[i - 1] + gridspacing_coalesce;
        }

        for (int i = 0; i < DUNROLLX; i++) {
            float r2 = (dx[i] * dx[i] + dyz2) * atom.w;
            float tmp = exp2f(-r2);
            densityvalx[i] += tmp;
            tmp *= invisovalue;
            densitycolx[i].x += tmp * color.x;
            densitycolx[i].y += tmp * color.y;
            densitycolx[i].z += tmp * color.z;
        }
    }

    for (int i = 0; i < DUNROLLX; i++) {
        densitygrid[outaddr + i * DBLOCKSZX] += densityvalx[i];
        voltexmap[outaddr + i * DBLOCKSZX].x += densitycolx[i].x;
        voltexmap[outaddr + i * DBLOCKSZX].y += densitycolx[i].y;
        voltexmap[outaddr + i * DBLOCKSZX].z += densitycolx[i].z;
    }
}