#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult(int* results, int* data, int* vec) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int result_val = 0;
    // Utilize shared memory for 'vec' to minimize global memory access latency
    __shared__ int shared_vec[cuda_features];

    // Load 'vec' into shared memory
    if (threadIdx.x < cuda_features) {
        shared_vec[threadIdx.x] = vec[threadIdx.x];
    }
    __syncthreads();

    // Calculate result using shared memory
    for (int i = 0; i < cuda_features; i++) {
        result_val += shared_vec[i] * data[(index * cuda_features) + i];
    }
    results[index] = result_val;
}