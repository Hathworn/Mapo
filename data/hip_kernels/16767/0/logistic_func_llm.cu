#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// FILE IO RELATED
#define MAX_ROWS_TRAINING 16896
#define MAX_COLUMNS_TRAINING 26
#define MAX_ROWS_TESTING 4096
#define MAX_COLUMNS_TESTING 26
#define MAX_CHAR 300

__constant__ int features = 26;
__constant__ int num_rows = 16896;

long mem_cpy_time = 0;
long beta_cpy_time = 0;

__global__ void logistic_func(float* log_func_v, float* betas, float* data) {
    int row_index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Utilize shared memory to reduce global memory access
    __shared__ float shared_betas[MAX_COLUMNS_TRAINING];
    if (threadIdx.x < features) {
        shared_betas[threadIdx.x] = betas[threadIdx.x];
    }
    __syncthreads();
    
    // Check if row_index is within bounds
    if (row_index < num_rows) {
        float temp = 0.0f;
        for (int j = 0; j < features; j++) {
            float accessed_data = data[(row_index * features) + j];
            temp += shared_betas[j] * accessed_data;
        }
        log_func_v[row_index] = 1.0f / (1.0f + expf(-temp));
    }
}