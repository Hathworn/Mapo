#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//FILE IO RELATED
//max number of lines in the training dataset
#define MAX_ROWS_TRAINING 16896
// max number of columns/features in the training dataset
#define MAX_COLUMNS_TRAINING 26
// max number of rows in the testing dataset
#define MAX_ROWS_TESTING 4096
// max number of columns in the testing data
#define MAX_COLUMNS_TESTING 26
//max number of characters/line
#define MAX_CHAR 300

__constant__ int features = 26;
__constant__ int num_rows = 16896;

long mem_cpy_time = 0;
long beta_cpy_time = 0;

// parallelized across the rows

// parallelized across the features

__global__ void log_gradient(float* log_func_v, float* gradient, float* betas, float* data, int* yvec) {
    extern __shared__ float sdata[];  // Shared memory for intermediate results
    int feature_index = blockIdx.x * blockDim.x + threadIdx.x;
    float temp = 0.0f;

    // Reduce global memory access by reading to shared memory
    for (int i = threadIdx.x; i < num_rows; i += blockDim.x) {
        float sub = log_func_v[i] - yvec[i];
        sdata[threadIdx.x] = sub * data[(i * features) + feature_index];
        __syncthreads();

        // Reduce within the block
        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                sdata[threadIdx.x] += sdata[threadIdx.x + s];
            }
            __syncthreads();
        }
    }

    // Write the result for this block to global memory
    if (threadIdx.x == 0) {
        atomicAdd(&gradient[feature_index], sdata[0]);
    }
}