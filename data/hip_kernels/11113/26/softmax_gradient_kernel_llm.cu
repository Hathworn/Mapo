#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    // Offset pointers to this block's data
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;

    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    // Initialize shared memory for reduction
    reduction_buffer[idx] = 0.0f;

    // Partial reduction in global memory
    float sum = 0.0f;
    for (int i = idx; i < dim; i += blockDim.x) {
        sum += dY[i] * Y[i];
    }
    reduction_buffer[idx] = sum;
    __syncthreads();

    // Single-threaded reduction of shared memory
    if (idx == 0) {
        sum = 0.0f;
        for (int i = 0; i < blockDim.x; ++i) {
            sum += reduction_buffer[i];
        }
        reduction_buffer[0] = sum;
    }
    __syncthreads();

    // Calculate gradient using the final reduction result
    float inner_product = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - inner_product);
    }
}