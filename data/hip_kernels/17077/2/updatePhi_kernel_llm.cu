#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updatePhi_kernel(int n, bool* d_flags, float* d_energy, float* d_fatigue, float theta) {
    // Calculate global index for the current thread
    unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
    // Loop through all elements with a stride equal to the total number of threads
    for (unsigned int stride = blockDim.x * gridDim.x; index < n; index += stride) {
        // Use ternary operator directly for simplicity
        d_flags[index] = (d_energy[index] - d_fatigue[index]) > theta;
    }
}