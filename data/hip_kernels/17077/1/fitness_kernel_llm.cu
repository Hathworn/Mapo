#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fitness_kernel(int* chromosome, int* collision) {
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    int temp = chromosome[bid];
    extern __shared__ int cache[]; // Shared memory for intermediate results

    // Parallel initialization
    cache[tid] = 0;
    if (tid < bid) {
        int d = abs(temp - chromosome[tid]);
        cache[tid] = (d == 0 || d == (bid - tid)) ? 1 : 0;
    }
    __syncthreads();

    // Efficient reduction using loop unrolling
    for (unsigned int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (tid < i) {
            cache[tid] += cache[tid + i];
        }
        __syncthreads();
    }

    // Store the result into global memory
    if (tid == 0) {
        atomicAdd(collision, cache[0]);
    }
}