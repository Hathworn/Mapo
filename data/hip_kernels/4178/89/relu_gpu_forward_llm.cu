#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Use shared memory for reduction of global memory access
    extern __shared__ float shared_mem[];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    if (tid < N) {
        shared_mem[local_tid] = in[tid];
        __syncthreads(); // Ensure all threads have loaded their data

        if (shared_mem[local_tid] > 0) {
            out[tid] = shared_mem[local_tid];
        } else {
            out[tid] = 0;
        }
    }
}