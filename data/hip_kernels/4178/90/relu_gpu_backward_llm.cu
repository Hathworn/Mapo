#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Utilizing fast math intrinsic for improved performance
    if (tid < N) ingrad[tid] = __fmul_rn(__int2float_rn(indata[tid] > 0), outgrad[tid]);
}