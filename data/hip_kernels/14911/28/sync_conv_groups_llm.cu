#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Utilize __syncthreads() for actual sync at block level
__global__ void sync_conv_groups() {
    __syncthreads();  // Synchronize threads within each block
}