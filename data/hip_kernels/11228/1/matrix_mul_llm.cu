#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for matrix multiplication
__global__ void matrix_mul(float* d_A, float* d_B, float* d_C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns)
{
    __shared__ float ds_M[TILE_WIDTH][TILE_WIDTH];
    __shared__ float ds_N[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int Row = by*TILE_WIDTH + ty;
    int Col = bx*TILE_WIDTH + tx;

    float Pvalue = 0;

    // Loop over tiles
    for (int m = 0; m < (numAColumns-1)/TILE_WIDTH+1; ++m) 
    {
        // Load data into shared memory with boundary checks
        ds_M[ty][tx] = (Row < numARows && m*TILE_WIDTH+tx < numAColumns) ? d_A[Row*numAColumns + m*TILE_WIDTH+tx] : 0;
        ds_N[ty][tx] = (Col < numBColumns && m*TILE_WIDTH+ty < numBRows) ? d_B[(m*TILE_WIDTH+ty)*numBColumns+Col] : 0;

        // Synchronize threads to ensure data is loaded
        __syncthreads();

        // Compute matrix multiplication on tile
        #pragma unroll // Unroll loop for better performance
        for (int k = 0; k < TILE_WIDTH; ++k)
        {
            Pvalue += ds_M[ty][k] * ds_N[k][tx];
        }

        // Synchronize before loading new data into shared memory
        __syncthreads();
    }

    // Write result to global memory
    if (Row < numCRows && Col < numCColumns)
    {
        d_C[Row*numCColumns+Col] = Pvalue;
    }
}