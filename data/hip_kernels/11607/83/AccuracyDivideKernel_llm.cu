#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Calculate unique index for each thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure only one thread executes the division to avoid race conditions
    if (idx == 0) {
        *accuracy /= N;
    }
}