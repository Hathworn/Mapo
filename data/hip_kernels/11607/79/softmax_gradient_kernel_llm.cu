#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0;

    // Unroll loop for better performance
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Improve reduction with better synchronization and unrolling
    if (idx == 0) {
        for (int i = 1; i < blockDim.x; ++i) {
            reduction_buffer[0] += reduction_buffer[i];
        }
    }
    __syncthreads();

    tmp = reduction_buffer[0];
    // Compute gradient with optimized memory access
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}