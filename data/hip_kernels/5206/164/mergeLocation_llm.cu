#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Use shared memory for faster access
    __shared__ short2 shared_loc[256]; 
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Load data to shared memory
        shared_loc[threadIdx.x] = loc_[ptidx];
        __syncthreads();

        // Scale coordinates
        short2 loc = shared_loc[threadIdx.x];
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}