#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    // Compute normalized thread index for better utilization
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Use FMA instructions for faster computation
    float cx = __fmaf_rn(u[flow_row_offset + j], time_scale, (float)j + 1.0f);
    float cy = __fmaf_rn(v[flow_row_offset + j], time_scale, (float)i + 1.0f);

    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];
    float weight;

    // Inline atomicAdd calls for pixel updates without branches
    if ((tx < w) && (tx >= 0) && (ty < h) && (ty >= 0)) {
        weight = dx * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    tx -= 1;
    if ((tx < w) && (tx >= 0) && (ty < h) && (ty >= 0)) {
        weight = (1.0f - dx) * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    ty -= 1;
    if ((tx < w) && (tx >= 0) && (ty < h) && (ty >= 0)) {
        weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    tx += 1;
    if ((tx < w) && (tx >= 0) && (ty < h) && (ty >= 0)) {
        weight = dx * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}