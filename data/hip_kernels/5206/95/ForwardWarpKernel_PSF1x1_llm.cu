#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Use 1D index for better memory coalescing and more uniform workload distribution
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  
    int totalThreads = gridDim.x * blockDim.x;

    for (int idx_i = idx; idx_i < w * h; idx_i += totalThreads) {
        int j = idx_i % w;
        int i = idx_i / w;
        
        int flow_row_offset = i * flow_stride;
        int image_row_offset = i * image_stride;

        float u_ = u[flow_row_offset + j];
        float v_ = v[flow_row_offset + j];

        // Bottom left corner of target pixel
        float cx = u_ * time_scale + (float)j + 1.0f;
        float cy = v_ * time_scale + (float)i + 1.0f;
        // Pixel containing bottom left corner
        int tx = __float2int_rn(cx);
        int ty = __float2int_rn(cy);

        float value = src[image_row_offset + j];
        // Fill pixel
        if (!(tx >= w || tx < 0 || ty >= h || ty < 0)) {
            atomicAdd(dst + ty * image_stride + tx, value);  // Use atomicAdd outside of condition
        }
    }
}