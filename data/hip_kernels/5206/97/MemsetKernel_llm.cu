#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Optimize memory access pattern
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Optimize memory access pattern

    if (i < h && j < w) // Use logical AND for conditional check
    {
        const int pos = i * w + j;
        image[pos] = value;
    }
}