#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_MonoBlock(unsigned char *buffer, long size, unsigned int *histo) {
    __shared__ unsigned int temp[256];

    // Initialize shared memory to zero
    temp[threadIdx.x] = 0;
    __syncthreads();

    // Calculate the index based on grid size for larger arrays
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = gridDim.x * blockDim.x;

    // Loop over the buffer with an increment of the total number of threads
    while (globalIndex < size) {
        atomicAdd(&temp[buffer[globalIndex]], 1);
        globalIndex += offset;
    }

    __syncthreads();

    // Accumulate the results into global histogram
    atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
}