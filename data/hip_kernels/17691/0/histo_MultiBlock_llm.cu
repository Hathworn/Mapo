#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define SIZE 2048*1024
#define BLOCKS 1000
#define THREADS 256

__global__ void histo_MultiBlock( unsigned char *buffer, long size, unsigned int *histo ) {

    // Declare shared memory with initial zeroing
    __shared__ unsigned int temp[256];
    if (threadIdx.x < 256) {
        temp[threadIdx.x] = 0;
    }
    __syncthreads();

    // Calculate initial index and stride
    int i = threadIdx.x + blockIdx.x * THREADS;
    int offset = THREADS * BLOCKS;
    
    // Accumulate local histogram in shared memory
    while (i < size) {
        atomicAdd(&temp[buffer[i]], 1);
        i += offset;
    }
    __syncthreads();

    // Combine results into the global histogram
    if (threadIdx.x < 256) {
        atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
    }
}