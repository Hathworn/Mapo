#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_copy_rc_cu_z(const double *din, double *zout, const int n) {
    // Optimize loop with shared memory and avoid bank conflicts
    extern __shared__ double shared[];
    const int igpt =
    (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;

    if (igpt < n) {
        shared[threadIdx.x] = din[igpt]; // Load data into shared memory
        __syncthreads(); // Synchronize threads
        const int offset = 2 * igpt;
        zout[offset] = shared[threadIdx.x]; // Write into global memory
        zout[offset + 1] = 0.0e0;
    }
}