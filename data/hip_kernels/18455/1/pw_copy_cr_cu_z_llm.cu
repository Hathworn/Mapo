#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_copy_cr_cu_z(const double * __restrict__ zin, double * __restrict__ dout, const int n) {
    // Calculate the global thread index
    int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Ensure not to exceed the array bounds
    if (igpt < n) {
        // Copy every alternate element from zin to dout
        dout[igpt] = zin[2 * igpt];
    }
}