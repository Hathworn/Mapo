#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_scatter_cu_z(double *c, const double *pwcc, const double scale, const int ngpts, const int nmaps, const int *ghatmap) {
    // Determine global thread ID
    const int igpt = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;

    // Check if thread index is within bounds
    if (igpt < ngpts) {
        const int map_idx = ghatmap[igpt];
        const double pwcc_val_real = scale * pwcc[2 * igpt];
        const double pwcc_val_imag = scale * pwcc[2 * igpt + 1];

        // Map scaled real and imaginary parts
        c[2 * map_idx] = pwcc_val_real;
        c[2 * map_idx + 1] = pwcc_val_imag;

        if (nmaps == 2) {
            const int map_idx_ngpts = ghatmap[igpt + ngpts];
            // Map additional scaled real and negated imaginary parts
            c[2 * map_idx_ngpts] = pwcc_val_real;
            c[2 * map_idx_ngpts + 1] = -pwcc_val_imag;
        }
    }
}