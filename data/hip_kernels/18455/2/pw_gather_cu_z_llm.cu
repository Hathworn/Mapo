#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_gather_cu_z(double *pwcc, const double *c, const double scale, const int ngpts, const int *ghatmap) {

    // Calculate global thread index using threadIdx, blockIdx, and blockDim
    int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Ensure that the thread operates only within bounds
    if (igpt < ngpts) {
        int index = 2 * ghatmap[igpt];
        // Optimize memory access pattern 
        pwcc[2 * igpt] = scale * c[index];
        pwcc[2 * igpt + 1] = scale * c[index + 1];
    }
}