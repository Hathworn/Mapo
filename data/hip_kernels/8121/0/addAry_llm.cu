#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// CUDA-C includes

extern "C" void runCudaPart();

// Optimized kernel function
__global__ void addAry(int *ary1, int *ary2)
{
    int indx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    ary1[indx] += ary2[indx]; // Perform addition at global index
}