#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flipCuda(const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height, bool horizontal, bool vertical)
{
    // Calculate input coordinates using improved indexing
    const uint32_t inX = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t inY = blockIdx.y * blockDim.y + threadIdx.y;

    // Bounds check for in-bounds coordinates
    if (inX < width && inY < height) {
        // Conditional assignment for output coordinates
        const uint32_t outX = horizontal ? (width - 1 - inX) : inX;
        const uint32_t outY = vertical ? (height - 1 - inY) : inY;

        // Efficient memory access pattern
        out[outY * rowSizeOut + outX] = in[inY * rowSizeIn + inX];
    }
}