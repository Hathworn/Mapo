#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rotateCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, float inXStart, float inYStart, uint32_t width, uint32_t height, float cosAngle, float sinAngle)
{
    // Calculate output pixel coordinates
    uint32_t outX = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t outY = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if the thread is within output image bounds
    if (outX < width && outY < height)
    {
        // Compute exact input coordinates using inverse rotation
        const float exactInX = inXStart + cosAngle * outX + sinAngle * outY;
        const float exactInY = inYStart - sinAngle * outX + cosAngle * outY;
        
        // Convert to integer coordinates for input image
        const int32_t inX = static_cast<int32_t>(exactInX);
        const int32_t inY = static_cast<int32_t>(exactInY);

        // Update output pointer index
        out += outY * rowSizeOut + outX;

        // Validate input pixel is within bounds for bilinear interpolation
        if (inX >= 0 && inX < width - 1 && inY >= 0 && inY < height - 1)
        {
            // Update input pointer index
            in += inY * rowSizeIn + inX;

            // Calculate interpolation weights
            const float probX = exactInX - inX;
            const float probY = exactInY - inY;

            // Perform bilinear interpolation and round the result
            const float mean = *in * (1.0f - probX) * (1.0f - probY) +
                               *(in + 1) * probX * (1.0f - probY) +
                               *(in + rowSizeIn) * (1.0f - probX) * probY +
                               *(in + rowSizeIn + 1) * probX * probY +
                               0.5f;

            // Write result to output
            *out = static_cast<uint8_t>(mean);
        }
        else
        {
            // Set output to zero if outside valid input zone
            *out = 0;
        }
    }
}