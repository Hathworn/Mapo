#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void invertCuda( const uint8_t * __restrict__ in, uint32_t rowSizeIn, uint8_t * __restrict__ out, uint32_t rowSizeOut, uint32_t width, uint32_t height )
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if ( x < width && y < height ) {
        // Use shared memory for efficient memory access
        __shared__ uint8_t tile[32][32];

        // Load input data into shared memory
        uint32_t linearIndex = y * rowSizeIn + x;
        tile[threadIdx.y][threadIdx.x] = in[linearIndex];

        __syncthreads(); // Ensure all threads have loaded data

        // Write inverted data to output
        if (x < width && y < height) {
            out[y * rowSizeOut + x] = ~tile[threadIdx.y][threadIdx.x];
        }
    }
}