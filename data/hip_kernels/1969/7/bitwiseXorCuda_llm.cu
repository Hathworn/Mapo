#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitwiseXorCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate unique thread index
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Use single memory access calculation per iteration
    if (x < width && y < height) {
        const uint32_t globalIdx = y * width + x;
        out[globalIdx] = in1[globalIdx] ^ in2[globalIdx];
    }
}