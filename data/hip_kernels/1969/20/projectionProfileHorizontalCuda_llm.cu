#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void projectionProfileHorizontalCuda(const uint8_t * image, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * projection)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockIdx.y;  // Optimize: Use only y-dimension for block index

    // Iterate over all elements in the column, handling multiple rows per thread
    if (x < width) {
        uint32_t sum = 0;
        for (uint32_t i = y * blockDim.y + threadIdx.y; i < height; i += blockDim.y * gridDim.y) {
            const uint8_t * imageX = image + i * rowSize + x;
            sum += *imageX;
        }

        // Optimize: Use atomicAdd after processing all rows to reduce contention
        atomicAdd(&projection[x], sum);
    }
}