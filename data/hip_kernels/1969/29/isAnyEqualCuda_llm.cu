#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void isAnyEqualCuda(const uint8_t *image, uint8_t *value, size_t valueCount, uint32_t width, uint32_t height, uint32_t *differenceCount)
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x < width && y < height) {
        const uint32_t id = y * width + x;

        // Use shared memory to store values
        __shared__ uint8_t sharedValue[256];
        if (threadIdx.x < valueCount) {
            sharedValue[threadIdx.x] = value[threadIdx.x];
        }
        __syncthreads();

        bool equal = false;

        // Compare with values in shared memory
        for (uint32_t i = 0; i < valueCount; ++i) {
            if (image[id] == sharedValue[i]) {
                equal = true;
                break;
            }
        }

        // Use atomicAdd to update count safely
        if (equal) {
            atomicAdd(differenceCount, 1);
        }
    }
}