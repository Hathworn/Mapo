#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyCuda(const uint8_t *in, uint32_t rowSizeIn, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height) {
    // Calculate linear index for 1D access
    const uint32_t idx = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t idy = blockDim.y * blockIdx.y + threadIdx.y;

    // Combine 2D coordinates into a linear index for optimization
    const uint32_t globalIdx = idy * rowSizeOut + idx;
    
    if (idx < width && idy < height) {
        out[globalIdx] = in[idy * rowSizeIn + idx];
    }
}