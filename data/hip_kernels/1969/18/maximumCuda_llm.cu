#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void maximumCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height) {
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if the thread is within bounds
    if (x < width && y < height) {
        // Calculate index directly using linear index to optimize pointer access
        const uint32_t index = y * width + x;
        const uint8_t *in1X = in1 + y * rowSizeIn1 + x;
        const uint8_t *in2X = in2 + y * rowSizeIn2 + x;
        uint8_t *outX = out + y * rowSizeOut + x;

        // Use conditional operator for maximum value assignment
        (*outX) = max(*in1X, *in2X);
    }
}