#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bitwiseOrCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height) 
{
    // Calculate combined thread index
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        uint32_t idIn1 = y * rowSizeIn1 + x;
        uint32_t idIn2 = y * rowSizeIn2 + x;
        uint32_t idOut = y * rowSizeOut + x;
        
        // Perform bitwise OR operation
        out[idOut] = in1[idIn1] | in2[idIn2];
    }
}