#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void isEqualCuda( const uint8_t * in1, uint32_t rowSizeIn1, const uint8_t * in2, uint32_t rowSizeIn2, uint32_t width, uint32_t height, uint32_t * isEqual )
{
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint32_t y = blockDim.y * blockIdx.y + threadIdx.y;

    // Early exit for threads outside the bounds
    if (x >= width || y >= height) return;

    const uint32_t partsEqual = static_cast<uint32_t>( in1[y * rowSizeIn1 + x] == in2[y * rowSizeIn2 + x] );
    
    // Use shared memory to aggregate results within a block
    __shared__ uint32_t sharedEqual[32]; // Assuming warp size is 32
    sharedEqual[threadIdx.x] = partsEqual;

    __syncthreads();

    // Reduction within the block
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        if (threadIdx.x < offset) {
            sharedEqual[threadIdx.x] &= sharedEqual[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Only one thread per block performs atomic operation
    if (threadIdx.x == 0) {
        atomicAnd(isEqual, sharedEqual[0]);
    }
}