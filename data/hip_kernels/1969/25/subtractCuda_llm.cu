#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subtractCuda(const uint8_t *in1, uint32_t rowSizeIn1, const uint8_t *in2, uint32_t rowSizeIn2, uint8_t *out, uint32_t rowSizeOut, uint32_t width, uint32_t height)
{
    // Calculate global indices
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure within bounds
    if (x < width && y < height) {
        // Inline calculation for improved performance
        const uint8_t in1Val = *(in1 + y * rowSizeIn1 + x);
        const uint8_t in2Val = *(in2 + y * rowSizeIn2 + x);
        uint8_t *outX = out + y * rowSizeOut + x;
        *outX = (in1Val > in2Val) ? (in1Val - in2Val) : 0;
    }
}