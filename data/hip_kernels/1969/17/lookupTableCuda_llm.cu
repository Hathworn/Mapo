#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lookupTableCuda(const uint8_t * in, uint32_t rowSizeIn, uint8_t * out, uint32_t rowSizeOut, uint32_t width, uint32_t height, const uint8_t * table)
{
    // Calculate global thread ID
    const uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure threads are within the matrix dimensions
    if (x < width && y < height) {
        // Load input value, access lookup table, and store in output
        const uint8_t value = in[y * rowSizeIn + x];
        out[y * rowSizeOut + x] = table[value];
    }
}