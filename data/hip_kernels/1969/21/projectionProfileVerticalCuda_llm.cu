#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void projectionProfileVerticalCuda(const uint8_t * image, uint32_t rowSize, uint32_t width, uint32_t height, uint32_t * projection) 
{
    // Calculate the x-coordinate for threads, y is block index
    const uint32_t x = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Iterate over rows for the current x coordinate
    if (x < width) {
        for (uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; y < height; y += blockDim.y * gridDim.y) {
            const uint8_t * imageY = image + y * rowSize + x;
            atomicAdd(&projection[y], *imageY); // Atomic add to avoid race condition
        }
    }
}