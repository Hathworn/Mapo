#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mat_mult_kernel(int *a, int *b, int *c, int mat_rows, int mat_cols) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for improved coalescing

    while (tid < mat_rows) {
        int res = 0;
        for (int i = 0; i < mat_cols; i++) {
            res += a[tid * mat_cols + i] * b[i]; // Accumulate results
        }
        c[tid] = res; // Store result in global memory
        tid += stride; // Use calculated stride for next position
    }
}