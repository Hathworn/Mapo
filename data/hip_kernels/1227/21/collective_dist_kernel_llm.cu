#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double get_collective_dist(int *dist, int rows, int cols, int col) {
    double sum = 0;
    for (int i = 0; i < rows; i++) {
        if (dist[i * cols + col] == 0) {
            return 0;
        }
        sum += (1 / (double)dist[i * cols + col]);
    }
    return sum;
}

__global__ void collective_dist_kernel(int *dist, int rows, int cols, double *col_dist)
{
    // Compute global thread id for unique thread indexing
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop for better memory coalescing and data access pattern
    for (int col = tid; col < cols; col += blockDim.x * gridDim.x) {
        col_dist[col] = get_collective_dist(dist, rows, cols, col);
    }
}