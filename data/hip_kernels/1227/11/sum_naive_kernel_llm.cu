#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_naive_kernel(int *arr, int size, int *sum) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory for local sums
    __shared__ int local_sum;
    if (threadIdx.x == 0) local_sum = 0;
    __syncthreads();

    // Use stride loop improving spatial locality
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        atomicAdd(&local_sum, arr[i]);
    }
    __syncthreads();

    // Atomic add to global sum from one thread per block
    if (threadIdx.x == 0) atomicAdd(sum, local_sum);
}