#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void block_sum_kernel(int *arr, int size, int *block_sums) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int my_sum = 0;
    __shared__ int smem[128];

    // Use block-level instead of grid-level striding for better occupancy
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        my_sum += arr[i];
    }
    smem[threadIdx.x] = my_sum;

    // Barrier then use parallel reduction to get block sum
    __syncthreads();
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {  // Use bit shift for division by 2
        if (threadIdx.x < i) {
            smem[threadIdx.x] += smem[threadIdx.x + i];
        }
        __syncthreads();
    }
    // Block sum added to global arr
    if (threadIdx.x == 0) {
        block_sums[blockIdx.x] = smem[0];
    }
}