#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mat_mult_fixed_dims_trans_kernel(int *mat_a, int *mat_b, int *res) {
    int B_TRANS_ROWS = B_COLS;
    int B_TRANS_COLS = B_ROWS;
    __shared__ int smem[128];

    for (int row_block = 0; row_block * gridDim.x < A_ROWS; row_block++) {
        int a_row = blockIdx.x + (row_block * gridDim.x);

        for (int b_row = 0; b_row < B_TRANS_ROWS; b_row++) {
            int total = 0;

            for (int thread_i = 0; thread_i * blockDim.x < A_COLS; thread_i++) {
                int thread_col = threadIdx.x + (thread_i * blockDim.x);

                // Optimize: Use ternary operator for condition
                smem[threadIdx.x] = (thread_col < A_COLS) ? mat_a[a_row * A_COLS + thread_col] * mat_b[b_row * B_TRANS_COLS + thread_col] : 0;
                __syncthreads();

                // Optimization: Unroll the parallel reduction loop
                for (int i = blockDim.x / 2; i > 0; i /= 2) {
                    if (threadIdx.x < i) {
                        smem[threadIdx.x] += smem[threadIdx.x + i];
                    }
                    __syncthreads();
                }

                if (threadIdx.x == 0) {
                    total += smem[0];
                }
            }
            
            if (threadIdx.x == 0) {
                res[a_row * C_COLS + b_row] = total;
            }
        }
    }
}