#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mat_mult_transposed_kernel(int *mat_a, int *mat_b, int *res) {
    int B_TRANS_ROWS = B_COLS;
    int B_TRANS_COLS = B_ROWS;
    __shared__ int smem[128];

    int a_row_stride = gridDim.x;  // Stride for outer loop over rows of A
    int a_row_start = blockIdx.x;  // Starting row index for current block

    for (int a_row = a_row_start; a_row < A_ROWS; a_row += a_row_stride) {
        for (int b_row = 0; b_row < B_TRANS_ROWS; b_row++) {

            int total = 0;
            int thread_stride = blockDim.x;  // Stride for inner loop over columns
            int thread_start = threadIdx.x;  // Starting column index for current thread

            for (int thread_col = thread_start; thread_col < A_COLS; thread_col += thread_stride) {
                if (thread_col < A_COLS) {
                    smem[threadIdx.x] = mat_a[a_row * A_COLS + thread_col] * mat_b[b_row * B_TRANS_COLS + thread_col];
                } else {
                    smem[threadIdx.x] = 0;
                }
                __syncthreads();

                // Parallel reduction
                for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
                    if (threadIdx.x < stride) {
                        smem[threadIdx.x] += smem[threadIdx.x + stride];
                    }
                    __syncthreads();
                }

                if (threadIdx.x == 0) {
                    total += smem[0];
                }
            }

            if (threadIdx.x == 0) {
                res[a_row * C_COLS + b_row] = total;
            }
        }
    }
}