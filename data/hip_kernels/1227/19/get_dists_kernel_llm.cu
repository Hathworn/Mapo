#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void get_dists_kernel(const int * beg_pos, const int* adj_list, const int * weights, bool * mask, int* dists, int * update_dists, const int num_vtx) {

    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    
    // Ensure we have valid threads to process vertices
    if (tid < num_vtx) {
    
        // Process only when the mask is true
        if (mask[tid] == true) {
            mask[tid] = false;  // Reset mask immediately
            
            int start = beg_pos[tid];
            int end = beg_pos[tid + 1];
            
            // Iterate over all edges of the current vertex
            for (int edge = start; edge < end; edge++) {
                int other = adj_list[edge];
                
                int new_dist = dists[tid] + weights[edge];
                
                // Atomic operation to safely update distance
                atomicMin(&update_dists[other], new_dist);
            }
        }
    }
}