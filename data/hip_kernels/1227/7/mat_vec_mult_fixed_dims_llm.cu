#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mat_vec_mult_fixed_dims(int *mat, int *vec, int *res) {
    const int mat_rows = 1024;
    const int mat_cols = 512;
    // Use dynamic shared memory for more flexibility
    extern __shared__ int smem[];

    for (int block_i = 0; block_i * gridDim.x < mat_rows; block_i++) {
        int row = blockIdx.x + (block_i * gridDim.x);

        if (row < mat_rows) {
            int row_total = 0;
            for (int thread_i = 0; thread_i * blockDim.x < mat_cols; thread_i++) {
                int col = threadIdx.x + (thread_i * blockDim.x);
                if (col < mat_cols) {
                    // Load mult in shared memory
                    smem[threadIdx.x] = mat[row * mat_cols + col] * vec[col];
                } else {
                    smem[threadIdx.x] = 0;  // Handle out-of-bounds access
                }
                __syncthreads();

                // Parallel reduction with optimized memory access
                for (int i = blockDim.x / 2; i > 0; i /= 2) {
                    if (threadIdx.x < i) {
                        smem[threadIdx.x] += smem[threadIdx.x + i];
                    }
                    __syncthreads();
                }

                // Only 1 thread needs to update the row total
                if (threadIdx.x == 0)
                    row_total += smem[0];
            }

            // Store result
            if (threadIdx.x == 0)
                res[row] = row_total;
        }
    }
}