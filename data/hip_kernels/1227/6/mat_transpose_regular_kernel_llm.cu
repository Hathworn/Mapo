#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mat_transpose_regular_kernel(int *mat, int *res) {
    // Define constants
    const int tile_dim = 32;
    const int blocks_per_row = 32;
    const int rows_per_block_iter = 64;

    // Calculate tile row and column
    int tile_row = blockIdx.x / blocks_per_row;
    int tile_col = blockIdx.x % blocks_per_row;

    // Calculate intra-tile row and column
    int intile_row = threadIdx.x / tile_dim;
    int intile_col = threadIdx.x % tile_dim;

    // Calculate global row and column
    int my_row_base = tile_row * tile_dim + intile_row;
    int my_col_base = tile_col * tile_dim + intile_col;

    // Prefetch data using registers to reduce global memory accesses
    int shared_mat[tile_dim]; // Store tile in shared memory

    // Loop over block iterations - unroll the loop for better performance
    #pragma unroll
    for (int block_iter = 0; block_iter < 16; block_iter++) {
        int my_row = my_row_base + (rows_per_block_iter * block_iter);
        int my_col = my_col_base;
        // Load data into shared memory
        shared_mat[intile_row] = mat[(my_row * 1024) + my_col];

        // Synchronize threads
        __syncthreads();

        // Write the transposed data to the output
        res[(my_col * 1024) + my_row] = shared_mat[intile_row];

        // Synchronize threads
        __syncthreads();
    }
}