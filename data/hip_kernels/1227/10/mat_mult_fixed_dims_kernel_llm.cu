#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mat_mult_fixed_dims_kernel(int *mat_a, int *mat_b, int *res) {
    // Shared memory allocation for partial results
    __shared__ int smem[128]; 
    int a_row;
    
    // Traverse rows of matrix A
    for (int row_block = 0; row_block * gridDim.x < A_ROWS; row_block++) {
        a_row = blockIdx.x + (row_block * gridDim.x);
        
        // Traverse columns of matrix B
        for (int b_col = 0; b_col < B_COLS; b_col++) {
            int thread_col;
            smem[threadIdx.x] = 0;  // Initialize shared memory for each thread

            // Perform matrix multiplication and store in shared memory
            for (int thread_i = 0; thread_i * blockDim.x < A_COLS; thread_i++) {
                thread_col = threadIdx.x + (thread_i * blockDim.x);
                if (thread_col < A_COLS) { // Skip load and multiply if out of bounds
                    smem[threadIdx.x] += mat_a[a_row * A_COLS + thread_col] * mat_b[thread_col * B_COLS + b_col];
                }
                __syncthreads();
            }

            // Parallel reduction to compute sum of products
            for (int i = blockDim.x / 2; i > 0; i /= 2) {
                if (threadIdx.x < i) {
                    smem[threadIdx.x] += smem[threadIdx.x + i];
                }
                __syncthreads();
            }

            // Write the result to the output matrix
            if (threadIdx.x == 0) {
                res[a_row * C_COLS + b_col] = smem[0];
            }
        }
    }
}