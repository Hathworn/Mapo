#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mat_mult_kernel(int *mat_a, int *mat_b, int *result, int a_rows, int a_cols, int b_cols) {
    // Compute row and column for the current thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < a_rows && col < b_cols) {
        int temp_res = 0;
        for (int k = 0; k < a_cols; k++) {
            temp_res += mat_a[row * a_cols + k] * mat_b[k * b_cols + col];
        }
        result[row * b_cols + col] = temp_res;
    }
}