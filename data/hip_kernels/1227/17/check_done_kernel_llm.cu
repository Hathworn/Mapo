#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void check_done_kernel(bool *mask, int num_vtx, bool *finished) {

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    bool local_finished = true; // Local variable to reduce global memory access

    while (tid < num_vtx) {
        if (mask[tid]) {
            local_finished = false;
            break; // Exit loop early if condition is met
        }
        tid += blockDim.x * gridDim.x;
    }

    // Perform reduction across all threads in the block
    __shared__ bool block_finished;
    if (threadIdx.x == 0) {
        block_finished = true;
    }
    __syncthreads();

    atomicAnd(&block_finished, local_finished);
    __syncthreads();

    if (threadIdx.x == 0 && !block_finished) {
        *finished = false;
    }
}