#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_sssp_data(bool * d_mask, int* d_dists, int* d_update_dists, const int source, const int num_vtx) {
    // Calculate thread id
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Process only if tid is valid
    if (tid < num_vtx) {
        bool is_source = (source == tid);
        d_mask[tid] = is_source;
        int initial_dist = is_source ? 0 : INT_MAX;
        d_dists[tid] = initial_dist;
        d_update_dists[tid] = initial_dist;
    }
}