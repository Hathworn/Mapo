#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_dists_kernel(const int *beg_pos, const int *adj_list, const int *weights, bool *mask, int *dists, int *update_dists, const int num_vtx) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < num_vtx) {
        int current_dist = update_dists[tid];  // Cache 'update_dists[tid]' for reuse
        if (dists[tid] > current_dist) {
            dists[tid] = current_dist;         // Update 'dists' only if needed
            mask[tid] = true;                  // Mark the mask as true
        }
        update_dists[tid] = dists[tid];        // Update 'update_dists' with new distance
    }
}