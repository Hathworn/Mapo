#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mat_transpose_coalesced_kernel(int *mat, int *res) {
    // Define constants
    const int tile_dim = 32;
    const int blocks_per_row = 32;
    const int rows_per_block_iter = 64;

    // Shared memory for tiles
    __shared__ int smem[tile_dim * tile_dim];

    // Compute tile position
    int tile_row = blockIdx.x / blocks_per_row;
    int tile_col = blockIdx.x % blocks_per_row;

    // Compute thread position within tile
    int intile_row = threadIdx.x / tile_dim;
    int intile_col = threadIdx.x % tile_dim;

    // Loop over multiple block iterations
    for (int block_iter = 0; block_iter < 16; block_iter++) {
        // Compute global read position
        int read_row = tile_row * tile_dim + intile_row + block_iter * rows_per_block_iter;
        int read_col = tile_col * tile_dim + intile_col;

        // Load tile into shared memory
        smem[intile_row * tile_dim + intile_col] = mat[read_row * 1024 + read_col];
        __syncthreads();

        // Compute global write position
        int write_row = tile_col * tile_dim + intile_row;
        int write_col = tile_row * tile_dim + intile_col + block_iter * rows_per_block_iter;

        // Write transposed tile to output
        res[write_row * 1024 + write_col] = smem[intile_col * tile_dim + intile_row];
    }
}