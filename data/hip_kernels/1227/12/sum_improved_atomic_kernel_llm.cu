#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_improved_atomic_kernel(int *arr, int size, int *sum) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int my_sum = 0;

    // Calculate the thread's assigned range with stride
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        my_sum += arr[i];
    }

    // Use shared memory to store the local sum of each thread
    extern __shared__ int smem[];
    smem[threadIdx.x] = my_sum;
    __syncthreads();

    // Parallel reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            smem[threadIdx.x] += smem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Atomic addition to the global sum
    if (threadIdx.x == 0) {
        atomicAdd(sum, smem[0]);
    }
}