#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multiplyGlobal(unsigned const* left, unsigned const* right, unsigned* result, size_t size)
{
    // Calculate unique thread index for row and column
    auto row = blockIdx.y * blockDim.y + threadIdx.y;
    auto col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check boundaries
    if (row < size && col < size) {
        unsigned sum = 0; // Use unsigned to avoid automatic type conversion

        // Use shared memory to reduce global memory access
        __shared__ unsigned sharedLeft[blockDim.y][blockDim.x];
        __shared__ unsigned sharedRight[blockDim.y][blockDim.x];

        for (int tile = 0; tile < size / blockDim.x + (size % blockDim.x == 0 ? 0 : 1); tile++) {
            if (row < size && tile * blockDim.x + threadIdx.x < size) {
                sharedLeft[threadIdx.y][threadIdx.x] = left[row * size + tile * blockDim.x + threadIdx.x];
            } else {
                sharedLeft[threadIdx.y][threadIdx.x] = 0;
            }

            if (col < size && tile * blockDim.y + threadIdx.y < size) {
                sharedRight[threadIdx.y][threadIdx.x] = right[(tile * blockDim.y + threadIdx.y) * size + col];
            } else {
                sharedRight[threadIdx.y][threadIdx.x] = 0;
            }

            __syncthreads(); // Synchronize threads to ensure all data is loaded into shared memory

            for (int k = 0; k < blockDim.x; k++) {
                sum += sharedLeft[threadIdx.y][k] * sharedRight[k][threadIdx.x];
            }

            __syncthreads(); // Make sure all calculations are complete before loading next tile
        }

        result[row * size + col] = sum; // Write result to global memory
    }
}