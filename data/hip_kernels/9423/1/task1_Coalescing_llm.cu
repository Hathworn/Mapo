#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void task1_Coalescing(unsigned const* a, unsigned const* b, unsigned* result, size_t size)
{
    // Calculate index and check bounds
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        // Perform multiplication for coalesced access
        result[index] = a[index] * b[index];
    }
}