#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void task2Kernel(unsigned const* a, unsigned const* b, unsigned* result, size_t size)
{
    // Calculate global index for threads
    auto index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop to handle all elements beyond block size
    for (; index < size; index += blockDim.x * gridDim.x) {
        result[index] = a[index] * b[index];
    }
}