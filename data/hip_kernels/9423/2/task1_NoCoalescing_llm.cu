#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void task1_NoCoalescing(unsigned const* a, unsigned const* b, unsigned* result, size_t size)
{
    // Use shared memory for better memory access pattern
    extern __shared__ unsigned shared_a[];
    extern __shared__ unsigned shared_b[];

    auto tid = threadIdx.x;
    auto index = blockIdx.x * blockDim.x + tid;

    if (index < size) {
        shared_a[tid] = a[index];
        shared_b[tid] = b[index];
        __syncthreads(); // Synchronize threads to ensure shared memory is ready

        result[index] = shared_a[tid] * shared_b[tid];
    }
}