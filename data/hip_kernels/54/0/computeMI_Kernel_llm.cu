#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t calcCuda(float *output, const uint8_t *input, const size_t *size);

static hipDeviceProp_t deviceProperties_;
__global__ void computeMI_Kernel(float *MIs, uint8_t *input, int rowCount, int colCount, int *countNZ, int offset)
{
    int globalId = blockIdx.x * blockDim.x + threadIdx.x + offset;
    if (globalId >= rowCount * (rowCount - 1) / 2) return;

    // Calculate pair (a, b) from globalId
    int a = 1, b = 0, i = globalId;
    for (int j = rowCount - 1; j > 1; j--) {
        if (i < j) break;
        a++;
        i -= j;
    }
    b = i; i = globalId - (a * (a - 1)) / 2;

    int joints[2][2] = {0};  // Initialize joint probability array
    int countNZA = countNZ[a], countNZB = countNZ[b];
    
    MIs += globalId;
    *MIs = 0;

    uint8_t *inputA = input + a, *inputB = input + b;
    // Use registers to store row-wise data
    for (int k = 0; k < colCount; k++, inputA += rowCount, inputB += rowCount) {
        joints[*inputA][*inputB]++;
    }

    // Precompute inverse of colCount for optimization
    float colCountInv = 1.0f / colCount;

    // Compute mutual information
    for (int k = 0; k < 4; k++) {
        int localA = k % 2, localB = k / 2;
        float joint = joints[localA][localB];

        if (joint != 0) {
            joint *= colCountInv;
            int localCountA = localA ? countNZA : colCount - countNZA;
            int localCountB = localB ? countNZB : colCount - countNZB;
            *MIs += joint * log2f(joint / ((float)localCountA * colCountInv) / ((float)localCountB * colCountInv));
        }
    }
}