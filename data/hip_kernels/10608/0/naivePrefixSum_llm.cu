#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Device input vectors
int *d_a;
// Device output vector
int *d_b

__global__ void naivePrefixSum(int *A, int *B, int size, int iteration) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for faster access
    extern __shared__ int sharedB[];
    if (index < size) {
        sharedB[threadIdx.x] = B[index];
        __syncthreads();

        // Check and compute prefix sum
        if (threadIdx.x >= (1 << (iteration - 1)))
            A[index] = sharedB[threadIdx.x] + sharedB[threadIdx.x - (1 << (iteration - 1))];
        else
            A[index] = sharedB[threadIdx.x];
    }
}