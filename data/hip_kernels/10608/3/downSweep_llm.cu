#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Device input vectors
int *d_a;
// Device output vector
int *d_b;

__global__ void downSweep(int *A, int size, int iteration) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < size) {
        int aux;
        int offset = 1 << (iteration + 1); // Precompute the offset value for reuse
        int targetIdx = index + 1; // Compute target index for readability
        if (targetIdx % offset == 0) { // Use precomputed offset 
            int swapIdx = index - (1 << iteration); // Compute swap index once
            aux = A[swapIdx];
            A[swapIdx] = A[index];
            A[index] = aux + A[index];
        }
    }
}