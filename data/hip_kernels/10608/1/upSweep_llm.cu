#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Device input vectors
int *d_a;
// Device output vector
int *d_b;

__global__ void upSweep(int *A, int size, int iteration) {
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll condition checks to reduce branch divergence
    int step = 1 << (iteration + 1);
    if (index < size && (index & (step - 1)) == step - 1) {
        A[index] += A[index - (1 << iteration)];
    }
}