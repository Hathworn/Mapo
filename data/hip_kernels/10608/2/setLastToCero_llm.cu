#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Device input vectors
int *d_a;
// Device output vector
int *d_b;

__global__ void setLastToCero(int *A, int size) {
    // Ensure only one thread at last index executes
    if (threadIdx.x == 0 && blockIdx.x == (size - 1) / blockDim.x) {
        A[size - 1] = 0;
    }
}