#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convn_full_kernel(float *output, float *data, float *kernel, int H, int W, int kH, int kW) {
    // Matrix index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // fH, fW stands for full H and full W
    const int fH = H + kH - 1;
    const int fW = W + kW - 1;

    if (x >= fH || y >= fW) 
        return;

    // Initialize sum
    float sum = 0;

    // Use shared memory for kernel
    __shared__ float sharedKernel[1024]; // Assuming kH * kW <= 1024

    // Load kernel to shared memory
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    if (tid < kH * kW) {
        sharedKernel[tid] = kernel[tid];
    }
    __syncthreads();

    // Convolution loop simplified with shared memory
    for (int i = 0; i < kW; ++i) {
        for (int j = 0; j < kH; ++j) {
            int ii = y - i;
            int jj = x - j;

            // Check boundary conditions
            if (ii >= 0 && ii < W && jj >= 0 && jj < H) {
                sum += sharedKernel[i * kH + j] * data[ii * H + jj];
            }
        }
    }

    // Update output
    output[y * fH + x] = sum;
}