#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convn_valid_kernel(float *output, float *data, float *kernel, const int H, const int W, const int kH, const int kW) {
    // Matrix index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // vH, vW stands for valid H and valid W
    const int vH = H - kH + 1;
    const int vW = W - kW + 1;

    if (x >= vH || y >= vW)
        return;
    
    // Update x, y temporarily using shared memory for better performance
    __shared__ float sh_kernel[1024]; // Shared memory for kernel
    __shared__ float sh_data[1024];   // Shared memory for data
    
    // Load kernel and data into shared memory
    for (int i = threadIdx.x; i < kH * kW; i += blockDim.x) {
        sh_kernel[i] = kernel[i];
    }
    
    for (int j = threadIdx.y; j < H * W; j += blockDim.y) {
        sh_data[j] = data[j];
    }
    
    __syncthreads();

    float sum = 0;
    for (int i = 0; i < kW; ++i)
        for (int j = 0; j < kH; ++j)
            sum += sh_kernel[i * kH + j] * sh_data[(y + j) * W + (x + i)];

    output[y * vH + x] = sum;
}