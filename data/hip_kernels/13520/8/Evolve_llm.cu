#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Evolve(int *val, int *aux, int n) {
    // Cache block and thread indices
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure threads are within valid range
    if (i > 0 && i < n - 1 && j > 0 && j < n - 1) {
        // Calculate flat index for the center
        int centerIndex = i * n + j;

        // Access neighboring elements
        int up = val[centerIndex - n];                          // (i-1, j)
        int upright = val[centerIndex - n + 1];                 // (i-1, j+1)
        int right = val[centerIndex + 1];                       // (i, j+1)
        int rightdown = val[centerIndex + n + 1];               // (i+1, j+1)
        int down = val[centerIndex + n];                        // (i+1, j)
        int downleft = val[centerIndex + n - 1];                // (i+1, j-1)
        int left = val[centerIndex - 1];                        // (i, j-1)
        int leftup = val[centerIndex - n - 1];                  // (i-1, j-1)

        // Sum the values of all neighbors
        int sum = up + upright + right + rightdown + down + downleft + left + leftup;

        // Determine new state based on rules
        int estado = val[centerIndex];
        if (sum == 3) {
            estado = 1;  // Birth
        } else if (estado == 1 && (sum < 2 || sum > 3)) {
            estado = 0;  // Death
        }

        // Write new state to auxiliary array
        aux[centerIndex] = estado;
    }
}
```
