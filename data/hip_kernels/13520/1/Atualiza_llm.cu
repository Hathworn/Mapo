#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define tam 1.0
#define dx 0.00001
#define dt 0.000001
#define T 0.01
#define kappa 0.000045

__global__ void Atualiza(double *u, double *u_prev, const int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory to improve memory access efficiency
    extern __shared__ double shared_u_prev[];
    
    // Load data into shared memory
    if (idx <= n) {
        shared_u_prev[threadIdx.x] = u_prev[idx];
    }
    __syncthreads();

    if (idx == 0) {
        u[0] = u[n] = 0.; // enforce boundary condition
    }
    else if (idx < n) {
        // Calculate using shared memory
        u[idx] = shared_u_prev[threadIdx.x] + kappa * dt / (dx * dx) *
                 (shared_u_prev[threadIdx.x - 1] - 2 * shared_u_prev[threadIdx.x] + shared_u_prev[threadIdx.x + 1]);
    }
}