#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define tam 1.0
#define dx 0.00001
#define dt 0.000001
#define T 0.01
#define kappa 0.000045

__global__ void Inicializacao(double *uprev, const int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate the position only if within valid index range for optimization.
    if (idx <= n) {
        double x = idx * dx;
        // Use ternary operator for cleaner condition assignment.
        uprev[idx] = (x <= 0.5) ? (200 * x) : (200 * (1.0 - x));
    }
}