#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Random( float *results, long int n, unsigned int seed ) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
hiprandState_t state;

hiprand_init(seed, blockIdx.x, 0, &state);
results[ idx ] = (float)hiprand(&state) / 1000.0f;
// if( idx < n ){
//   results[ idx ] = 1.0;
// }
}