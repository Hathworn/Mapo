#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Random(float *results, long int n, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {  // Ensure idx is within bounds
        hiprandState_t state;
        hiprand_init(seed, idx, 0, &state);  // Use idx for unique sequence per thread
        results[idx] = (float)hiprand(&state) / 1000.0f;
    }
}