#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Random(float *results, int n, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Check idx is within bounds to avoid out of bounds memory access
    if (idx < n) {
        // Use thread index for more varied random state initialization
        hiprandState_t state;
        hiprand_init(seed, idx, 0, &state);
        // Write output using the valid index
        results[idx] = hiprand(&state) / 1000.0f;
    }
}