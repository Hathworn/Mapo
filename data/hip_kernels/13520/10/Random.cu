#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Random( float *results, int n, unsigned int seed ) {
int idx = blockIdx.x * blockDim.x + threadIdx.x;
hiprandState_t state;

hiprand_init(seed, blockIdx.x, 0, &state);
results[ idx ] = hiprand(&state) / 1000.0f;
}