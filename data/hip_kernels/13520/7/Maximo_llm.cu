#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Maximo(double *input, double *results, int n) {
    extern __shared__ double sdata[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    double x = (idx < n) ? input[idx] : -DBL_MAX; // Initialize with minimum double value for reduction
    sdata[tx] = x; 
    __syncthreads();

    // Use warp shuffle reduction to improve performance in the last warp
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (tx < offset) {
            if (sdata[tx] < sdata[tx + offset]) {
                sdata[tx] = sdata[tx + offset];
            }
        }
        __syncthreads();
    }

    if (tx == 0) {
        results[blockIdx.x] = sdata[0];
    }
}