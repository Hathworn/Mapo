#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define tam 1.0
#define dx 0.00001
#define dt 0.000001
#define T 0.01
#define kappa 0.000045

__global__ void Maximo(double *input, double *results, int n) {
    extern __shared__ double sdata[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    // Initialize shared memory with zeros
    sdata[tx] = 0.0;
    
    // Load data into shared memory only if the index is valid
    if (idx < n) {
        sdata[tx] = input[idx];
    }
    __syncthreads();

    // Efficient reduction in shared memory
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (tx < offset && idx + offset < n) {
            if (sdata[tx] < sdata[tx + offset]) {
                sdata[tx] = sdata[tx + offset];
            }
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tx == 0) {
        results[blockIdx.x] = sdata[0];
    }
}