#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Somatorio(float *input, float *results, long int n) {
    extern __shared__ float sdata[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    // Initialize shared memory
    sdata[tx] = (idx < n) ? input[idx] : 0.0f;
    __syncthreads();

    // Perform reduction
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (tx < offset) {
            sdata[tx] += sdata[tx + offset];
        }
        __syncthreads(); // Synchronize within block
    }

    // Write result for this block to global memory
    if (tx == 0) {
        results[blockIdx.x] = sdata[0];
    }
}