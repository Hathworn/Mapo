#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel for matrix addition
__global__ void MatrixAdd_d(float *A, float *B, float *C, int N) {
    // Calculate the row and column index for the current thread
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Linear index computation
    int index = i * N + j;
    
    // Check boundaries and perform the addition
    if (i < N && j < N) {
        C[index] = A[index] + B[index];
    }
}