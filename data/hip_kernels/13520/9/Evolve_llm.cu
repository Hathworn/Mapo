#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Evolve( int *val, int *aux, int n ) {
    int sum = 0, estado;
    const int tx = threadIdx.x + 1, ty = threadIdx.y + 1;
    const int i = blockIdx.y * blockDim.y + threadIdx.y;
    const int j = blockIdx.x * blockDim.x + threadIdx.x;
    const int b2 = BSIZE + 2;
    __shared__ int sdata[b2][b2]; // Changed from float to int to match val and reduce memory use

    // Load center value
    if (i < n && j < n) {
        sdata[ty][tx] = val[i * n + j];
    }

    // Load halo cells
    if (threadIdx.x == 0 && j > 0) 
        sdata[ty][0] = val[i * n + j - 1]; // Left
    if (threadIdx.x == blockDim.x - 1 && j < n - 1) 
        sdata[ty][b2 - 1] = val[i * n + j + 1]; // Right
    if (threadIdx.y == 0 && i > 0) 
        sdata[0][tx] = val[(i - 1) * n + j]; // Up
    if (threadIdx.y == blockDim.y - 1 && i < n - 1) 
        sdata[b2 - 1][tx] = val[(i + 1) * n + j]; // Down

    // Load corners
    if (threadIdx.x == 0 && threadIdx.y == 0 && i > 0 && j > 0)
        sdata[0][0] = val[(i - 1) * n + j - 1]; // Left-Up corner
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0 && i > 0 && j < n - 1)
        sdata[0][b2 - 1] = val[(i - 1) * n + j + 1]; // Right-Up corner
    if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1 && i < n - 1 && j > 0)
        sdata[b2 - 1][0] = val[(i + 1) * n + j - 1]; // Left-Down corner
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1 && i < n - 1 && j < n - 1)
        sdata[b2 - 1][b2 - 1] = val[(i + 1) * n + j + 1]; // Right-Down corner

    __syncthreads();

    // Compute new state only for valid global indices
    if (i > 0 && i < n - 1 && j > 0 && j < n - 1) {
        estado = sdata[ty][tx];
        sum = sdata[ty - 1][tx] + sdata[ty - 1][tx + 1] + sdata[ty][tx + 1] +
              sdata[ty + 1][tx + 1] + sdata[ty + 1][tx] + sdata[ty + 1][tx - 1] +
              sdata[ty][tx - 1] + sdata[ty - 1][tx - 1];

        // Determine the state based on the sum
        if (sum == 3) {
            estado = 1;
        } else if ((estado == 1) && (sum < 2 || sum > 3)) {
            estado = 0;
        }
        aux[i * n + j] = estado;
    }
}