#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Utilize shared memory for improved memory access efficiency
__global__ void Atualiza(double *u, double *u_prev, const int n) {
    extern __shared__ double shared_u_prev[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tx = threadIdx.x;

    // Load data to shared memory
    if (idx < n) {
        shared_u_prev[tx] = u_prev[idx];
        if (tx == 0 && idx > 0) {
            shared_u_prev[tx - 1] = u_prev[idx - 1];
        }
        if (tx == blockDim.x - 1 && idx < n - 1) {
            shared_u_prev[tx + 1] = u_prev[idx + 1];
        }
    }
    __syncthreads();

    // Update u array with boundary condition check
    if (idx < n) {
        if (idx == 0) {
            u[0] = u[n] = 0.;  // Boundary condition
        } else {
            u[idx] = shared_u_prev[tx] + kappa * dt / (dx * dx) * 
                     (shared_u_prev[tx - 1] - 2 * shared_u_prev[tx] + shared_u_prev[tx + 1]);
        }
    }
}