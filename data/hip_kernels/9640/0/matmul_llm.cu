#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmul(float* A, float* B, float* C, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols) {
    // Initialize shared memory to store sub-matrices
    __shared__ float As[16][16];
    __shared__ float Bs[16][16];

    float CValue = 0;

    int Row = blockIdx.y * 16 + threadIdx.y;
    int Col = blockIdx.x * 16 + threadIdx.x;

    for (int k = 0; k < (ACols + 16 - 1) / 16; k++) {
        // Load subsets into shared memory with bounds checking
        if (Row < ARows && (k * 16 + threadIdx.x) < ACols) {
            As[threadIdx.y][threadIdx.x] = A[Row * ACols + k * 16 + threadIdx.x];
        } else {
            As[threadIdx.y][threadIdx.x] = 0.0;
        }

        if (Col < BCols && (k * 16 + threadIdx.y) < BRows) {
            Bs[threadIdx.y][threadIdx.x] = B[(k * 16 + threadIdx.y) * BCols + Col];
        } else {
            Bs[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads();  // Synchronize threads

        for (int n = 0; n < 16; ++n) {
            CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];
        }

        __syncthreads();  // Synchronize threads
    }

    // Write back the result if within bounds
    if (Row < CRows && Col < CCols) {
        C[Row * CCols + Col] = CValue;
    }
}