#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmul(float* A, float* B, float* C, int ARows, int ACols, int BRows, int BCols, int CRows, int CCols) {

    // Load tile dimensions into shared memory
    __shared__ float As[16][16];
    __shared__ float Bs[16][16];

    float CValue = 0;
    int Row = blockIdx.y * 16 + threadIdx.y;
    int Col = blockIdx.x * 16 + threadIdx.x;

    for (int k = 0; k < (ACols - 1) / 16 + 1; ++k) {

        // Collaborative loading of tiles into shared memory
        if (Row < ARows && k * 16 + threadIdx.x < ACols)
            As[threadIdx.y][threadIdx.x] = A[Row * ACols + k * 16 + threadIdx.x];
        else
            As[threadIdx.y][threadIdx.x] = 0.0f;

        if (k * 16 + threadIdx.y < BRows && Col < BCols)
            Bs[threadIdx.y][threadIdx.x] = B[(k * 16 + threadIdx.y) * BCols + Col];
        else
            Bs[threadIdx.y][threadIdx.x] = 0.0f;

        __syncthreads();

        // Matrix multiplication within block
        for (int n = 0; n < 16; ++n)
            CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

        __syncthreads();
    }

    // Store the result back to global memory
    if (Row < CRows && Col < CCols)
        C[Row * CCols + Col] = CValue;
}