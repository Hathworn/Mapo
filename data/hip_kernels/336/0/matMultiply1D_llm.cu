#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//#define NDEBUG

const static float eps = 1e-6;
const static size_t blocSize = 8;
const static size_t size = 1024;

__global__ void matMultiply1D(float* matA, float* matB, float* Dest, int dimensions)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < dimensions)
    {
        // Load a row of matA into shared memory to reduce global memory access.
        __shared__ float sharedMatA[blocSize][blocSize];

        for (int blockBase = 0; blockBase < dimensions; blockBase += blocSize)
        {
            // Load data from global mem to shared mem
            if (threadIdx.x + blockBase < dimensions)
            {
                sharedMatA[threadIdx.x][blockDim.x] = matA[i * dimensions + threadIdx.x + blockBase];
            }
            __syncthreads();

            for (int j = 0; j < dimensions; ++j)
            {
                float res = 0.0f;
                for (int k = 0; k < blocSize; ++k)
                {
                    if (k + blockBase < dimensions)
                    {
                        res += sharedMatA[k][threadIdx.x] * matB[(k + blockBase) * dimensions + j];
                    }
                }
                Dest[i * dimensions + j] = res;
            }
            __syncthreads();
        }
    }
}