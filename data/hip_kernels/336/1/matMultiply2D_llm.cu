#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//#define NDEBUG

const static float eps = 1e-6;
const static size_t blocSize = 8;
const static size_t size = 1024;

__global__ void matMultiply2D(float* matA, float* matB, float* Dest, int dimensions)
{
    // Shared memory for sub-matrices
    __shared__ float tileA[blocSize][blocSize];
    __shared__ float tileB[blocSize][blocSize];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int ix = tx + bx * blockDim.x;
    int iy = ty + by * blockDim.y;

    float res = 0.0f;
    // Loop over tiles
    for (int t = 0; t < (dimensions + blocSize - 1) / blocSize; ++t)
    {
        // Load tiles into shared memory
        if (ix < dimensions && (t * blocSize + ty) < dimensions)
            tileA[ty][tx] = matA[ix * dimensions + t * blocSize + ty];
        else
            tileA[ty][tx] = 0.0f;
        
        if (iy < dimensions && (t * blocSize + tx) < dimensions)
            tileB[ty][tx] = matB[(t * blocSize + tx) * dimensions + iy];
        else
            tileB[ty][tx] = 0.0f;

        __syncthreads(); // Synchronize to make sure all data is loaded
        
        // Compute product of tiles
        for (unsigned k = 0; k < blocSize; ++k)
        {
            res += tileA[ty][k] * tileB[k][tx];
        }
        
        __syncthreads(); // Synchronize before loading new tile
    }

    // Write the result to the destination matrix
    if (ix < dimensions && iy < dimensions)
    {
        Dest[ix * dimensions + iy] = res;
    }
}