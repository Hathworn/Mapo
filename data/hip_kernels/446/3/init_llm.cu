#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(unsigned int seed, hiprandState_t* states) {
    // Calculate unique ID for the thread
    unsigned int stateid = blockIdx.x * blockDim.x + threadIdx.x;

    /* Initialize the state with a unique sequence number for each thread */
    hiprand_init(seed, stateid, 0, &states[stateid]);
}