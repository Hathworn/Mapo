#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(unsigned int seed, hiprandState_t* states) {
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate a unique global index
    /* initialize the state with a unique sequence number for each thread */
    hiprand_init(seed, id, 0, &states[id]);
}