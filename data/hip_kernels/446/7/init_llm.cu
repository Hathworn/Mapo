#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(unsigned int seed, hiprandState_t* states) {
    // improved indexing for better performance
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // initialize the state using improved index to leverage more threads
    hiprand_init(seed, idx, 0, &states[idx]);
}