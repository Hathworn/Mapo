#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float machine_eps_flt() {
    typedef union {
        int i32;
        float f32;
    } flt_32;

    flt_32 s;
    s.f32 = 1.;
    s.i32++;
    return (s.f32 - 1.);
}

__device__ double machine_eps_dbl() {
    typedef union {
        long long i64;
        double d64;
    } dbl_64;

    dbl_64 s;
    s.d64 = 1.;
    s.i64++;
    return (s.d64 - 1.);
}

__global__ void calc_consts(float *fvals, double *dvals) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i == 0) {
        // Computing machine epsilon
        fvals[EPS] = machine_eps_flt();
        dvals[EPS] = machine_eps_dbl();

        float xf = 2.f, oldxf = 1.f;
        double xd = 2.0, oldxd = 1.0;

        // Calculating maximum representable value
        while (!isinf(xf)) {
            oldxf = xf;
            xf *= 2.f;
        }

        while (!isinf(xd)) {
            oldxd = xd;
            xd *= 2.0;
        }

        fvals[MAX] = oldxf;
        dvals[MAX] = oldxd;

        xf = 1.f, oldxf = 2.f;
        xd = 1.0, oldxd = 2.0;

        // Calculating minimum representable value
        while (xf != 0.f) {
            oldxf = xf;
            xf /= 2.f;
        }

        while (xd != 0.0) {
            oldxd = xd;
            xd /= 2.0;
        }

        fvals[MIN] = oldxf;
        dvals[MIN] = oldxd;
    }
}