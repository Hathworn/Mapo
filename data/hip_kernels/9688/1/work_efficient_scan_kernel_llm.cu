#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS 256
#define BLOCKS 32
#define NUM THREADS*BLOCKS

int seed_var =1239;

__global__ void work_efficient_scan_kernel(int *X, int *Y, int InputSize)
{
    extern __shared__ int XY[];
    int i= blockIdx.x*blockDim.x + threadIdx.x;
    
    // Load elements into shared memory
    if (i < InputSize) {
        XY[threadIdx.x] = X[i];
    }
    __syncthreads();

    // Forward inclusive scan
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        int index = (threadIdx.x + 1) * 2 * stride - 1;
        if (index < blockDim.x) {
            XY[index] += XY[index - stride];
        }
        __syncthreads();
    }

    // Reverse scan to distribute additions
    for (int stride = blockDim.x / 4; stride > 0; stride /= 2) {
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < blockDim.x) {
            XY[index + stride] += XY[index];
        }
        __syncthreads();
    }

    // Copy result to output array
    if (i < InputSize) {
        Y[i] = XY[threadIdx.x];
    }
    __syncthreads();

    // Handle block scan additions
    if (threadIdx.x == 0 && blockIdx.x > 0) {
        Y[i] += Y[blockIdx.x * blockDim.x - 1];
    }
    __syncthreads();
}