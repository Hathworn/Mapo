#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
    unsigned int i = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int ixj = i ^ j;

    // Ensure threads with the required ids are doing sorting
    if (ixj > i) {
        bool ascending = (i & k) == 0;
        // Use a single conditional block to improve efficiency
        if ((ascending && dev_values[i] > dev_values[ixj]) || (!ascending && dev_values[i] < dev_values[ixj])) {
            // Swap elements
            float temp = dev_values[i];
            dev_values[i] = dev_values[ixj];
            dev_values[ixj] = temp;
        }
    }
}