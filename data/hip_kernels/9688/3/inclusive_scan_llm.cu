#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void inclusive_scan(const unsigned int *X, unsigned int *Y, int N)
{
    extern __shared__ unsigned int XY[];

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input into shared memory
    if (i < N)
    {
        XY[threadIdx.x] = X[i];
    }
    __syncthreads();

    // Perform inclusive scan within block - optimized for shared memory access
    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) 
    {
        if (threadIdx.x >= stride)
        {
            XY[threadIdx.x] += XY[threadIdx.x - stride];
        }
        __syncthreads();
    }

    // Write result to global memory
    if (i < N)
    {
        Y[i] = XY[threadIdx.x];
    }
    __syncthreads();

    // Accumulate results from previous blocks - optimized for memory access
    if (blockIdx.x > 0 && threadIdx.x == 0)
    {
        unsigned int prevBlockSum = XY[blockDim.x - 1];
        for (int stride = 0; stride < blockIdx.x; stride++)
        {
            XY[threadIdx.x] += Y[stride * blockDim.x + blockDim.x - 1];
        }
        __syncthreads();

        // Update block results with accumulated prefix sum
        if (i < N)
        {
            Y[i] += prevBlockSum;
        }
    }
}