#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS 256
#define BLOCKS 32
#define NUM THREADS*BLOCKS

int seed_var = 1239;

__device__ void swap(int *xp, int *yp)
{
    int temp = *xp;
    *xp = *yp;
    *yp = temp;
}

__global__ void bitonic_sort_step(int *d_pr, int *d_bt, int j, int k)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int ixj = i ^ j;

    // Optimize: Use shared memory for better performance
    __shared__ int shared_d_pr[THREADS];
    __shared__ int shared_d_bt[THREADS];
    
    // Load data into shared memory
    shared_d_pr[threadIdx.x] = d_pr[i];
    shared_d_bt[threadIdx.x] = d_bt[i];
    __syncthreads();

    // Perform sorting using shared memory
    if (ixj > i)
    {
        if ((i & k) == 0)
        {
            if (shared_d_pr[threadIdx.x] > shared_d_pr[ixj % THREADS])
            {
                // Swap in shared memory
                swap(&shared_d_pr[threadIdx.x], &shared_d_pr[ixj % THREADS]);
                swap(&shared_d_bt[threadIdx.x], &shared_d_bt[ixj % THREADS]);
            }
        }
        else
        {
            if (shared_d_pr[threadIdx.x] < shared_d_pr[ixj % THREADS])
            {
                // Swap in shared memory
                swap(&shared_d_pr[threadIdx.x], &shared_d_pr[ixj % THREADS]);
                swap(&shared_d_bt[threadIdx.x], &shared_d_bt[ixj % THREADS]);
            }
        }
    }
    __syncthreads();

    // Store results back to global memory
    d_pr[i] = shared_d_pr[threadIdx.x];
    d_bt[i] = shared_d_bt[threadIdx.x];
}