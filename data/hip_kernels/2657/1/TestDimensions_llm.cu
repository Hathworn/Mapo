#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void TestDimensions()
{
    int const thread_id = blockIdx.x * blockDim.x +
                          blockIdx.y * blockDim.x * blockDim.y +
                          blockIdx.z * blockDim.x * blockDim.y * blockDim.z +
                          threadIdx.x +
                          threadIdx.y * blockDim.x +
                          threadIdx.z * blockDim.x * blockDim.y;

    // Simplified calculation for fixed printf calls for block and grid dimensions
    if (thread_id == 0)
    {
        printf("gridDim   = x: %6d / y: %6d / z: %6d\r\n", gridDim.x, gridDim.y, gridDim.z);
        printf("blockDim  = x: %6d / y: %6d / z: %6d\r\n", blockDim.x, blockDim.y, blockDim.z);
        printf("num_threads_per_block: %6d\r\n", blockDim.x * blockDim.y * blockDim.z);
        printf("num_blocks           : %6d\r\n", gridDim.x * gridDim.y * gridDim.z);
        printf("threads_stride       : %6d\r\n", blockDim.x * blockDim.y * blockDim.z * gridDim.x * gridDim.y * gridDim.z);
    }

    printf("tidx | %6d | %6d | %6d | bidx | %6d | %6d | %6d | "
           "gdim | %6d | %6d | %6d | bdim | %6d | %6d | %6d | "
           "thread_id | %6d |\r\n",
            threadIdx.x, threadIdx.y, threadIdx.z,
            blockIdx.x, blockIdx.y, blockIdx.z,
            gridDim.x, gridDim.y, gridDim.z,
            blockDim.x, blockDim.y, blockDim.z, thread_id);

    return;
}