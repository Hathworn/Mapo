#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcCDF(float *cdf, unsigned int *histo, int imageWidth, int imageHeight, int length) {
    // Use shared memory for fast access
    __shared__ float partialScan[SIZE_CDF];

    int i = threadIdx.x + blockDim.x * blockIdx.x;

    // Load histogram into shared memory and normalize
    if (i < SIZE_CDF && i < 256) {
        partialScan[i] = (float)histo[i] / (float)(imageWidth * imageHeight);
    }
    __syncthreads();

    // Hillis-Steele scan for prefix sum
    for (unsigned int stride = 1; stride <= SIZE_HISTO; stride *= 2) {
        unsigned int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index < SIZE_CDF && index < length) {
            partialScan[index] += partialScan[index - stride];
        }
        __syncthreads();
    }

    // Regularize the array using downsweep stage
    for (unsigned int stride = SIZE_HISTO / 2; stride > 0; stride /= 2) {
        __syncthreads();
        unsigned int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < SIZE_CDF && index + stride < length) {
            partialScan[index + stride] += partialScan[index];
        }
    }

    // Write the results back to the global memory
    __syncthreads();
    if (i < SIZE_CDF && i < 256) {
        cdf[i] = partialScan[i];  // Correct the output write
    }
}