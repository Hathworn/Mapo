#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcCDFnormalized(const unsigned int *histo, float *cdf, size_t width, size_t height) {
    // Shared memory for histogram accumulation to minimize global memory access
    __shared__ float tempCDF[1024];
    
    // Initialize shared memory for each thread
    tempCDF[threadIdx.x] = static_cast<float>(histo[threadIdx.x]);
    __syncthreads();

    // Inclusive scan to accumulate values in shared memory
    for (int offset = 1; offset < blockDim.x; offset *= 2) {
        float val = 0.0f;
        if (threadIdx.x >= offset) {
            val = tempCDF[threadIdx.x - offset];
        }
        __syncthreads();
        tempCDF[threadIdx.x] += val;
        __syncthreads();
    }

    // Normalize and write result back to global memory
    cdf[threadIdx.x] = tempCDF[threadIdx.x] * (1.0f / static_cast<float>(width * height));
}