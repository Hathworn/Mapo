#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int n, float a, float *x, float *y) {
    // Calculate index and stride for better utilization of threads
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use loop to handle larger data sizes
    for (int i = index; i < n; i += stride) {
        y[i] = a * x[i] + y[i];
    }
}