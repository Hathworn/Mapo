#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Efficient division using reciprocal
    float reciprocal = 1.0f / static_cast<float>(N);
    *accuracy *= reciprocal;
}