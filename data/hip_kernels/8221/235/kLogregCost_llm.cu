#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    // Launch only necessary threads
    if (tx >= numCases) return;

    const int label = int(labels[tx]);
    const float maxp = maxProbs[tx];
    const float labelp = probs[label * numCases + tx];

    labelLogProbs[tx] = __logf(labelp);

    // Early exit condition to improve performance
    if (labelp != maxp) {
        correctProbs[tx] = 0;
        return;
    }

    int numMax = 0;
    // Reduce iterations using shared memory
    __shared__ float sharedProbs[LOGREG_ERR_THREADS_X];
    for (int i = threadIdx.x; i < numOut; i += LOGREG_ERR_THREADS_X) {
        sharedProbs[threadIdx.x] = probs[i * numCases + tx];
        __syncthreads();
        if (sharedProbs[threadIdx.x] == maxp) numMax++;
        __syncthreads();
    }

    correctProbs[tx] = 1.0f / float(numMax);
}