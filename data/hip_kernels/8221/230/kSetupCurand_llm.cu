#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate the thread index once to improve efficiency.
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    
    // Initialize the random state for each thread without repeated calculation.
    hiprand_init(seed, tidx, 0, &state[tidx]);
}