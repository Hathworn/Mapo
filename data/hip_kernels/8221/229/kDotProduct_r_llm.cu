#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;  // Use register for partial sum

    // Unroll loop to improve memory access
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        sum += a[eidx] * b[eidx];
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Reduce phases
    for (uint stride = DP_BLOCKSIZE / 2; stride > 32; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x < 32) {
        volatile float* mysh = &shmem[threadIdx.x];
        mysh[0] += mysh[32];
        mysh[0] += mysh[16];
        mysh[0] += mysh[8];
        mysh[0] += mysh[4];
        mysh[0] += mysh[2];
        mysh[0] += mysh[1];
    }
    
    // Write result back to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}