#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float tmp = 0.0f;

    // Efficient partial reduction using warp-level primitives
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    if (idx < warpSize) {
        // Warp reduction within the first warp
        tmp = reduction_buffer[idx];
        for (int offset = warpSize / 2; offset > 0; offset /= 2) {
            tmp += __shfl_down(tmp, offset);
        }
        if (idx == 0) {
            reduction_buffer[0] = tmp;
        }
    }
    __syncthreads();
    
    // Compute gradient
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}