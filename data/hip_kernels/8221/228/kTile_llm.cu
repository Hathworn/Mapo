#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Flatten source image height and stride
    const uint srcStride = srcWidth;

    // Total number of threads
    const int numThreads = blockDim.x * gridDim.x;
    
    // Iterate over target indices this thread is responsible for
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Calculate target coordinate
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        
        // Compute corresponding source coordinates
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Map the pixel from source to target
        tgt[i] = src[srcY * srcStride + srcX];
    }
}
```
