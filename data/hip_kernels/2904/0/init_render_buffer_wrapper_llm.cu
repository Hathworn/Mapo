#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize by combining device and kernel function, removing unnecessary function call overhead
__global__ void init_render_buffer(int64_t* render_buffer, const uint32_t qw_count) {
    const uint32_t start = blockIdx.x * blockDim.x + threadIdx.x;
    const uint32_t step = blockDim.x * gridDim.x;
    for (uint32_t i = start; i < qw_count; i += step) {
        render_buffer[i] = EMPTY_KEY_64;
    }
}