#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_valid_pos(int32_t* pos_buff, int32_t* count_buff, const int32_t entry_count) {
    const int32_t start = threadIdx.x + blockDim.x * blockIdx.x;
    const int32_t step = blockDim.x * gridDim.x;

    // Use shared memory to cache count_buff values for better performance
    extern __shared__ int32_t shared_count_buff[];
    
    for (int32_t i = start; i < entry_count; i += step) {
        // Load count_buff into shared memory
        if (i < entry_count) {
            shared_count_buff[threadIdx.x] = count_buff[i];
        }
        __syncthreads(); // Ensure all threads have loaded their values

        // Optimize condition check and data writes
        if (VALID_POS_FLAG == pos_buff[i]) {
            pos_buff[i] = !i ? 0 : shared_count_buff[threadIdx.x - 1];
        }
    }
}