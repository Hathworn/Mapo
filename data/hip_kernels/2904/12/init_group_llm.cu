#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_group(int8_t* groups, const size_t group_count, const size_t col_count, const size_t* col_widths, const size_t* init_vals) {
    // Calculate the global thread index
    const auto thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Exit if thread index exceeds the group count
    if (thread_index >= group_count) {
        return;
    }
    
    // Loop through columns to initialize groups
    size_t group_offset = 0; // Accumulate offset
    for (size_t i = 0; i < col_count; ++i) {
        switch (col_widths[i]) {
        case 4:
            // Initialize 4-byte wide data
            *reinterpret_cast<uint32_t*>(groups + group_offset) = *reinterpret_cast<const uint32_t*>(init_vals + i);
            break;
        case 8:
            // Initialize 8-byte wide data
            reinterpret_cast<size_t*>(groups + group_offset)[thread_index] = init_vals[i];
            break;
        default:
            // No initialization if column width doesn't match expected cases
            break;
        }
        group_offset += col_widths[i] * group_count; // Update group offset
    }
}