#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void columnarize_groups(int8_t* columnar_buffer, const int8_t* rowwise_buffer, const size_t row_count, const size_t col_count, const size_t* col_widths, const size_t row_size) {
    // Calculate unique thread index in 1D within the grid.
    const auto thread_index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Exit if thread is out of bounds.
    if (thread_index >= row_count) return;
    
    auto read_ptr = rowwise_buffer + thread_index * row_size;
    auto col_base = columnar_buffer;
    
    // Iterate through each column.
    for (size_t i = 0; i < col_count; ++i) {
        // Use switch statement for different column widths
        switch (col_widths[i]) {
            case 8: {
                // Typecast pointers and perform write operation for width 8.
                int64_t* write_ptr = reinterpret_cast<int64_t*>(col_base) + thread_index;
                *write_ptr = *reinterpret_cast<const int64_t*>(read_ptr);
                break;
            }
            case 4: {
                // Typecast pointers and perform write operation for width 4.
                int32_t* write_ptr = reinterpret_cast<int32_t*>(col_base) + thread_index;
                *write_ptr = *reinterpret_cast<const int32_t*>(read_ptr);
                break;
            }
            default:;
        }
        // Move column base and read pointer as per current column's width.
        col_base += col_widths[i] * row_count;
        read_ptr += col_widths[i];
    }
}