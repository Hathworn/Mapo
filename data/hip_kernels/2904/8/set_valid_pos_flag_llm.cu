#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_valid_pos_flag(int32_t* pos_buff, const int32_t* count_buff, const int32_t entry_count) {
    const int32_t idx = threadIdx.x + blockDim.x * blockIdx.x;
    // Ensure the thread index does not surpass entry_count
    if (idx < entry_count && count_buff[idx]) { 
        pos_buff[idx] = VALID_POS_FLAG;
    }
}