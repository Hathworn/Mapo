#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iota(int const size, int *data, int const value)
{
    // Compute global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid-stride loop for improved utilization
    for (int i = idx; i < size; i += blockDim.x * gridDim.x)
    {
        data[i] = i + value;
    }
}