#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void DeviceMultiply(double* left, double* right, double* result, int left_rows, int left_cols, int right_cols) {
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Check if thread indices are within matrix bounds
    if (i < left_rows && j < right_cols) {
        double sum = 0.0;
        // Use registers instead of shared memory
        for (int k = 0; k < left_cols; ++k) {
            int left_idx = i * left_cols + k;
            int right_idx = k * right_cols + j;
            sum += left[left_idx] * right[right_idx];
        }
        int result_idx = i * right_cols + j;
        result[result_idx] = sum;
    }
}