#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth(float * __restrict__ v_new, const float * __restrict__ v) {
    // Calculate global index for the current thread
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;

    // Ensure thread does not access out-of-bounds memory
    if (myIdx < numThreads) {
        // Calculate indices of neighboring elements
        int myLeftIdx = max(0, myIdx - 1);
        int myRightIdx = min(numThreads - 1, myIdx + 1);

        // Fetch required elements from global memory once
        float myElt = v[myIdx];
        float myLeftElt = v[myLeftIdx];
        float myRightElt = v[myRightIdx];

        // Calculate smoothed value
        v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
    }
}