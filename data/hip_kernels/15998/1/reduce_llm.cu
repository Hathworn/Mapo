#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int *s) {
    int lane = threadIdx.x % warpSize;  // Get lane index in the warp
    int warpId = threadIdx.x / warpSize; // Get warp index

    s[threadIdx.x] = p;

    // Perform warp-level reduction using shuffle operations
    if (lane < 16) s[threadIdx.x] += s[threadIdx.x + 16];
    if (lane < 8) s[threadIdx.x] += s[threadIdx.x + 8];
    if (lane < 4) s[threadIdx.x] += s[threadIdx.x + 4];
    if (lane < 2) s[threadIdx.x] += s[threadIdx.x + 2];
    if (lane < 1) s[threadIdx.x] += s[threadIdx.x + 1];

    // Write warp result to shared memory
    if (lane == 0) {
        s[warpId] = s[threadIdx.x];
    }

    __syncthreads(); // Synchronize to ensure all warp reductions are complete

    // Reduce across warps
    if (warpId == 0) {
        if (lane < 16) s[lane] += s[lane + 16];
        if (lane < 8) s[lane] += s[lane + 8];
        if (lane < 4) s[lane] += s[lane + 4];
        if (lane < 2) s[lane] += s[lane + 2];
        if (lane < 1) s[lane] += s[lane + 1];
    }

    return s[0]; // Return the result from the first element
}

__global__ void reduce(unsigned int *d_out_shared, const unsigned int *d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    if (t == 0) {
        *d_out_shared = sr;
    }
}