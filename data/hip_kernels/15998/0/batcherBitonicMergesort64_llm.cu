#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid  = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    // Assuming input is 64 elements
    for (int stage = 0; stage <= 5; stage++)
    {
        int int_stage = 1 << stage;
        for (int substage = stage; substage >= 0; substage--)
        {
            int stride = 1 << substage;  // Calculate stride
            int pair = tid ^ stride;

            // Ensure valid pair and compare & swap
            if (pair > tid)
            {
                if ((tid & int_stage) == 0)
                {
                    if (sdata[tid] > sdata[pair])
                    {
                        float temp = sdata[tid];
                        sdata[tid] = sdata[pair];
                        sdata[pair] = temp;
                    }
                }
                else
                {
                    if (sdata[tid] < sdata[pair])
                    {
                        float temp = sdata[tid];
                        sdata[tid] = sdata[pair];
                        sdata[pair] = temp;
                    }
                }
            }
            __syncthreads();  // Synchronize threads
        }
    }

    d_out[tid] = sdata[tid];
}