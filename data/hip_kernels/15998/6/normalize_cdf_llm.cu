#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n) {
    // Calculate normalization constant once and store in shared memory
    __shared__ float normalization_constant;
    if (threadIdx.x == 0) {
        normalization_constant = 1.f / d_input_cdf[n - 1];
    }
    __syncthreads();
    
    // Compute global index
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform normalization if within bounds
    if (global_index_1d < n) {
        d_output_cdf[global_index_1d] = d_input_cdf[global_index_1d] * normalization_constant;
    }
}