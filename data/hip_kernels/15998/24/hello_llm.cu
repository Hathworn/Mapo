#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Store thread ID in a local variable for reuse
    int tid = threadIdx.x;
    
    // Use a single printf to reduce IO overhead
    printf("Hello world! I'm thread %d\n", tid);
}