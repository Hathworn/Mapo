#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use threadIdx.x to differentiate between threads within the block
    printf("Hello world! I'm a thread %d in block %d\n", threadIdx.x, blockIdx.x);
}