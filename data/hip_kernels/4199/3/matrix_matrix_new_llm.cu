#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized Global Kernel for Matrix-Matrix Multiplication
__global__ void matrix_matrix_new(int *a, int *b, int *c, int n_row, int n_col, int n_comm)
{
    // Calculate unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for common data to reduce global memory access
    __shared__ int shared_a[128];  // Assuming 128 threads per block, adjust as needed
    __shared__ int shared_b[128];

    while (tid < n_row * n_col)
    {
        // Determine row and column for current thread
        int i = tid / n_col;
        int j = tid % n_col;

        int temp = 0;
        // Iterate over the common dimension with tile optimization
        for (int k = 0; k < n_comm; k += blockDim.x)
        {
            // Load portions of a and b into shared memory
            if (k + threadIdx.x < n_comm) {
                shared_a[threadIdx.x] = a[i * n_comm + k + threadIdx.x];
                shared_b[threadIdx.x] = b[(k + threadIdx.x) * n_col + j];
            }
            __syncthreads();

            // Perform partial computation using shared memory
            for (int x = 0; x < blockDim.x; ++x)
            {
                temp += shared_a[x] * shared_b[x];
            }
            __syncthreads();
        }
        c[tid] = temp;
        tid += blockDim.x * gridDim.x;  // Process next element within the grid stride
    }
}