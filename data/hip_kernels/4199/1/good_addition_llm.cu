#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized vector addition kernel
__global__ void good_addition(int *a, int *b, int *c, int len) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;  // Calculate the stride
    for (int i = tid; i < len; i += stride) {  // Use stride loop to cover entire array
        c[i] = a[i] + b[i];  // Perform addition
    }
}