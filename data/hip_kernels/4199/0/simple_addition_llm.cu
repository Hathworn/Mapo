#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Optimized Vector addition kernel */

/*
Matrix Matrix multiplication with a single thread for each row
*/

/*
Matrix Matrix multiplication with a single thread for each result element
*/

/*
Matrix Vector multiplication with a block with 4 threads per block, shared block mem and parallel reduce
*/

__global__ void simple_addition(int *a, int *b, int *c, int len)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure tid is within bounds
    if (tid < len) {
        c[tid] = a[tid] + b[tid];
    }
}