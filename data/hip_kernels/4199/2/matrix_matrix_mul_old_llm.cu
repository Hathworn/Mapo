#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_matrix_mul_optimized(int *a, int *b, int *c, int n_row, int n_col, int n_comm)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Use 2D block and grid for row
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Use 2D block and grid for column

    if (row < n_row && col < n_col) {
        int sum = 0;
        for (int j = 0; j < n_comm; j++) {
            sum += a[row * n_comm + j] * b[j * n_col + col];
        }
        c[row * n_col + col] = sum;
    }
}