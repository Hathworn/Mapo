#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void weight_division_calc(float* sum_squared_afferent_values, float* afferent_weight_change_updater, float* weight_divisor, bool* neuron_in_plasticity_set, size_t total_number_of_neurons)
{
    // Global Index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Calculate the stride

    // Loop unrolling could be possible for more throughput
    for (; idx < total_number_of_neurons; idx += stride) {
        if (neuron_in_plasticity_set[idx]) {
            float sum_sq_value = sum_squared_afferent_values[idx];
            float weight_update = afferent_weight_change_updater[idx];
            if ((sum_sq_value - weight_update < 0.01))
                printf("NORMALIZATION DIFF VERY LARGE. DANGER OF SYNAPSES ALL -> ZERO");
            weight_divisor[idx] = sqrtf(sum_sq_value + weight_update) / sqrtf(sum_sq_value);
        }
    }
    // __syncthreads() is unnecessary as there's no shared memory used
}