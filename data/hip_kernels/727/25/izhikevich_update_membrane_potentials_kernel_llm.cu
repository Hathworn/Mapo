#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void izhikevich_update_membrane_potentials_kernel(float *d_membrane_potentials_v, float *d_states_u, float *d_param_a, float *d_param_b, float* d_current_injections, float* thresholds_for_action_potentials, float* last_spike_time_of_each_neuron, float* resting_potentials, float current_time_in_seconds, float timestep, size_t total_number_of_neurons) {

    // Convert timestep to milliseconds
    float eqtimestep = timestep * 1000.0f;
    // Get thread index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x; // Calculate stride for optimization

    while (idx < total_number_of_neurons) {
        // Compute updates for each neuron using Izhikevich equations
        float v = d_membrane_potentials_v[idx];
        float u = d_states_u[idx];
        float v_update = 0.04f * v * v + 5.0f * v + 140 - u + d_current_injections[idx];

        v += eqtimestep * v_update;
        u += eqtimestep * (d_param_a[idx] * (d_param_b[idx] * v - u));

        // Check for action potential threshold
        if (v >= thresholds_for_action_potentials[idx]) {
            v = resting_potentials[idx];
            last_spike_time_of_each_neuron[idx] = current_time_in_seconds;
        }

        // Intermediate results stored back
        d_membrane_potentials_v[idx] = v;
        d_states_u[idx] = u;

        // Move to the next neuron index
        idx += stride;
    }
    // Ensure all threads complete computations
    __syncthreads();
}