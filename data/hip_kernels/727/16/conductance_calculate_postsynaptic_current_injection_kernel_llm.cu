#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conductance_calculate_postsynaptic_current_injection_kernel( float* decay_term_values, float* reversal_values, int num_decay_terms, int* synapse_decay_values, float* neuron_wise_conductance_traces, float* d_neurons_current_injections, float * d_membrane_potentials_v, float timestep, size_t total_number_of_neurons){

    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Ensure no unnecessary branching by initializing current injection
    bool is_idx_valid = (idx < total_number_of_neurons);
    if (is_idx_valid) {
        d_neurons_current_injections[idx] = 0.0f;
        float membrane_potential_v = d_membrane_potentials_v[idx];

        // Loop unrolling and reduced memory access
        #pragma unroll
        for (int decay_id = 0; decay_id < num_decay_terms; decay_id++) {
            float synaptic_conductance_g = neuron_wise_conductance_traces[idx + decay_id*total_number_of_neurons];

            // Reduce exp computation overhead
            float decay_factor = expf(-timestep / decay_term_values[decay_id]);
            synaptic_conductance_g *= decay_factor;

            neuron_wise_conductance_traces[idx + decay_id*total_number_of_neurons] = synaptic_conductance_g;

            // Inline calculation of current injection
            d_neurons_current_injections[idx] += synaptic_conductance_g * (reversal_values[decay_id] - membrane_potential_v);
        }
    }
}