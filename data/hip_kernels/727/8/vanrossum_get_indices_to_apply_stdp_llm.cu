#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vanrossum_get_indices_to_apply_stdp (int* d_postsyns, float* d_last_spike_time_of_each_neuron, float* d_time_of_last_spike_to_reach_synapse, int* d_index_of_last_afferent_synapse_to_spike, bool* d_isindexed_ltd_synapse_spike, int* d_index_of_first_synapse_spiked_after_postneuron, float currtime, int* d_plastic_synapse_indices, size_t total_number_of_plastic_synapses) {
    int indx = threadIdx.x + blockIdx.x * blockDim.x;

    // Loop unrolling to improve performance
    int step = blockDim.x * gridDim.x;
    while (indx < total_number_of_plastic_synapses) {
        int idx = d_plastic_synapse_indices[indx];
        int postsynaptic_neuron = d_postsyns[idx];

        float spike_time = d_time_of_last_spike_to_reach_synapse[idx];
        // Combine conditional checks to minimize branching
        if (spike_time == currtime) {
            atomicExch(&d_index_of_last_afferent_synapse_to_spike[postsynaptic_neuron], idx);

            if (!d_isindexed_ltd_synapse_spike[postsynaptic_neuron]) {
                d_isindexed_ltd_synapse_spike[postsynaptic_neuron] = true;
                atomicExch(&d_index_of_first_synapse_spiked_after_postneuron[postsynaptic_neuron], idx);
            }
        }
        
        indx += step;
    }
}