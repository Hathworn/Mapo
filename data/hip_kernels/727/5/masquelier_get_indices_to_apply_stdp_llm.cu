#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void masquelier_get_indices_to_apply_stdp (
    int* d_postsyns, 
    float* d_last_spike_time_of_each_neuron, 
    float* d_time_of_last_spike_to_reach_synapse, 
    int* d_index_of_last_afferent_synapse_to_spike, 
    bool* d_isindexed_ltd_synapse_spike, 
    int* d_index_of_first_synapse_spiked_after_postneuron, 
    float currtime, 
    int* d_plastic_synapse_indices, 
    size_t total_number_of_plastic_synapses
) {
    int indx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;  // Calculate the stride

    // Loop through synapses with stride
    while (indx < total_number_of_plastic_synapses) {
        int idx = d_plastic_synapse_indices[indx];
        int postsynaptic_neuron = d_postsyns[idx];

        // Check if a synapse reached a neuron this timestep
        if (d_time_of_last_spike_to_reach_synapse[idx] == currtime) {
            atomicExch(&d_index_of_last_afferent_synapse_to_spike[postsynaptic_neuron], idx);
        }

        // Check if a synapse has fired and update if necessary
        if (!d_isindexed_ltd_synapse_spike[postsynaptic_neuron] && 
            d_time_of_last_spike_to_reach_synapse[idx] == currtime) {
            d_isindexed_ltd_synapse_spike[postsynaptic_neuron] = true;
            atomicExch(&d_index_of_first_synapse_spiked_after_postneuron[postsynaptic_neuron], idx);
        }

        // Move to the next index in the grid
        indx += stride;
    }
}