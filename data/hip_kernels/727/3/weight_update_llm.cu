#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weight_update(int* postsyn_neuron, bool* neuron_in_plasticity_set, float* current_weight, float* weight_divisor, int* d_plastic_synapse_indices, size_t total_number_of_plastic_synapses) {

    // Calculate global index
    int indx = threadIdx.x + blockIdx.x * blockDim.x;

    // Loop optimally through all synapses using grid stride
    for (; indx < total_number_of_plastic_synapses; indx += blockDim.x * gridDim.x) {
        int idx = d_plastic_synapse_indices[indx];
        int postneuron = postsyn_neuron[idx];

        // Check neuron plasticity status
        if (neuron_in_plasticity_set[postneuron]) {
            float division_value = weight_divisor[postneuron];

            // Avoid unnecessary division operations
            if (division_value != 1.0f) {
                current_weight[idx] /= division_value;
            }
        }
    }
}