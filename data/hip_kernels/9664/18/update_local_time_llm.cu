#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_local_time(int *next, double *local_time, double GTIME) {
    unsigned int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    int who = next[gtid];

    // Use branchless method to update local time when 'who' is valid
    if (who >= 0) {
        local_time[who] = GTIME;
    }
}