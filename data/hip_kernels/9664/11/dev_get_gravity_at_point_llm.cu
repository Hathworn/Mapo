#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dev_get_gravity_at_point( float eps2, float *eps, float *xh, float *yh, float *zh, float *xt, float *yt, float *zt, float *ax, float *ay, float *az, int n, float *field_m, float *fxh, float *fyh, float *fzh, float *fxt, float *fyt, float *fzt, int n_field) {
    // Use shared memory for reused values and reduce global memory accesses
    __shared__ float shared_eps2[1024]; 
    __shared__ float shared_field_m[1024];
    
    int tid = threadIdx.x + blockIdx.x * blockDim.x; // Optimized thread indexing
    int lane_id = threadIdx.x;

    // Preloading data into shared memory
    if (lane_id < n) {
        shared_eps2[lane_id] = eps[lane_id];
    }
    if (lane_id < n_field) {
        shared_field_m[lane_id] = field_m[lane_id];
    }
    __syncthreads();

    float dx, dy, dz, r2, tmp, dr2, eps2_total;

    if (tid < n) { // Ensure the thread operates only within range
        eps2_total = eps2 + shared_eps2[tid] * shared_eps2[tid];
        ax[tid] = 0; // Initialize acceleration components
        ay[tid] = 0;
        az[tid] = 0;

        for (int i = 0; i < n_field; i++) {
            dx = (fxh[i] - xh[tid]) + (fxt[i] - xt[tid]);
            dy = (fyh[i] - yh[tid]) + (fyt[i] - yt[tid]);
            dz = (fzh[i] - zh[tid]) + (fzt[i] - zt[tid]);
            dr2 = dx * dx + dy * dy + dz * dz;
            
            // Check to avoid division by zero
            if (dr2 > 0) {
                r2 = eps2_total + dr2;
                tmp = shared_field_m[i] / (r2 * sqrt(r2));
                ax[tid] += tmp * dx; // Accumulate acceleration
                ay[tid] += tmp * dy;
                az[tid] += tmp * dz;
            }
        }
    }
}