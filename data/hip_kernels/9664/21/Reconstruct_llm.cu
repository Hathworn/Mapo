#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Reconstruct(int *nex, unsigned long nextsize, double4 *pc, double4 *vc, double4 *a3, double4 *a, double4 *a1, double4 *a2, double4 *pva3, double4 *aaa) {
    unsigned int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Pre-calculate commonly used values
    int k = gtid / nextsize;
    int offset_gtid = gtid % nextsize;  // Offset within a particular nextsize segment
    int who = nex[offset_gtid];

    // Use a switch-case to minimize branching and improve readability
    switch (gtid / nextsize) {
        case 0:
            pc[who] = pva3[gtid];
            break;
        case 1:
            vc[who] = pva3[gtid];
            break;
        case 2:
            a3[who] = pva3[gtid];
            break;
        case 3:
            a[who] = aaa[offset_gtid];
            break;
        case 4:
            a1[who] = aaa[offset_gtid];
            break;
        case 5:
            a2[who] = aaa[offset_gtid];
            break;
    }
}