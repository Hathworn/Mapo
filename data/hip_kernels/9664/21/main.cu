#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "Reconstruct.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    int *nex = NULL;
hipMalloc(&nex, XSIZE*YSIZE);
unsigned long nextsize = 1;
double4 *pc = NULL;
hipMalloc(&pc, XSIZE*YSIZE);
double4 *vc = NULL;
hipMalloc(&vc, XSIZE*YSIZE);
double4 *a3 = NULL;
hipMalloc(&a3, XSIZE*YSIZE);
double4 *a = NULL;
hipMalloc(&a, XSIZE*YSIZE);
double4 *a1 = NULL;
hipMalloc(&a1, XSIZE*YSIZE);
double4 *a2 = NULL;
hipMalloc(&a2, XSIZE*YSIZE);
double4 *pva3 = NULL;
hipMalloc(&pva3, XSIZE*YSIZE);
double4 *aaa = NULL;
hipMalloc(&aaa, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
Reconstruct<<<gridBlock, threadBlock>>>(nex,nextsize,pc,vc,a3,a,a1,a2,pva3,aaa);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
Reconstruct<<<gridBlock, threadBlock>>>(nex,nextsize,pc,vc,a3,a,a1,a2,pva3,aaa);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
Reconstruct<<<gridBlock, threadBlock>>>(nex,nextsize,pc,vc,a3,a,a1,a2,pva3,aaa);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}