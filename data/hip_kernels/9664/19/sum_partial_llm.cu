#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_partial(double4 *a, double4 *b, unsigned int nextsize) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i >= nextsize) return;
    
    extern __shared__ double4 shaccelerations[];
    double4 myacc = b[i];
    
    // Load into shared memory: Threads in a warp can access efficiently
    shaccelerations[threadIdx.x] = a[i];
    __syncthreads(); // Ensure all threads have loaded their data into shared memory
    
    // Efficient use of shared memory data
    myacc.x += shaccelerations[threadIdx.x].x;
    myacc.y += shaccelerations[threadIdx.x].y;
    myacc.z += shaccelerations[threadIdx.x].z;
    
    b[i] = myacc;
}