#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void compute_potential_gpu(float *m, float *x, float *y, float *z, float *phi, int N, int N1) {
    // Calculate global thread index
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < (N1 == 0 ? N : N1)) {
        float xi = x[i];
        float yi = y[i];
        float zi = z[i];
        float potential = 0.0f;  // Initialize potential to zero

        // Loop through all points or from N1 to N when N1 is not zero
        for (int j = (N1 == 0 ? 0 : N1); j < N; j++) {
            if (i != j) {  // Ensure i and j are not the same
                // Calculate distance and potential contribution
                float rijx = xi - x[j];
                float rijy = yi - y[j];
                float rijz = zi - z[j];
                potential -= m[j] / sqrtf(rijx * rijx + rijy * rijy + rijz * rijz);
            }
        }
        phi[i] = potential;  // Store the calculated potential
    }
}