#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initvectors(double4 *acc3, float4 *apred) {
    // Calculate unique global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single statement to initialize double4 and float4
    acc3[i] = make_double4(0.0, 0.0, 0.0, acc3[i].w);
    apred[i] = make_float4(0.0f, 0.0f, 0.0f, apred[i].w);
}