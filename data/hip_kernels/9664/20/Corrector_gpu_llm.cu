#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Corrector_gpu(double GTIME, double *local_time, double *step, int *next, unsigned long nextsize, double4 *pos_CH, double4 *vel_CH, double4 *a_tot_D, double4 *a1_tot_D, double4 *a2_tot_D, double4 *a_H0, double4 *a3_H, double ETA6, double ETA4, double DTMAX, double DTMIN, unsigned int N) {

    unsigned int gtid = blockIdx.x * blockDim.x + threadIdx.x;

    if (gtid >= nextsize) return;  // Return early if thread ID is out of bounds

    // Load and precompute constants
    const int who = next[gtid];
    const int who1 = gtid + nextsize;
    const int who2 = who1 + nextsize;

    double h = GTIME - local_time[who];
    local_time[who] = GTIME;

    const double h1 = 0.5 * h;
    const double h2 = h1 * h1;
    const double h3 = 0.75 / (h1 * h1 * h1);
    const double h4 = 1.5 / (h2 * h2);
    const double h5 = 7.5 / (h2 * h2 * h1);
    const double over = 1.0 / 15.0;

    // Common calculations for each component
    for (int i = 0; i < 3; i++) {
        // Calculate magnitudes
        a_H0[gtid].w = a_H0[gtid][i] * a_H0[gtid][i] + a_H0[gtid][(i+1)%3] * a_H0[gtid][(i+1)%3] + a_H0[gtid][(i+2)%3] * a_H0[gtid][(i+2)%3];
        a_H0[who1].w = a_H0[who1][i] * a_H0[who1][i] + a_H0[who1][(i+1)%3] * a_H0[who1][(i+1)%3] + a_H0[who1][(i+2)%3] * a_H0[who1][(i+2)%3];
        a_H0[who2].w = a_H0[who2][i] * a_H0[who2][i] + a_H0[who2][(i+1)%3] * a_H0[who2][(i+1)%3] + a_H0[who2][(i+2)%3] * a_H0[who2][(i+2)%3];

        double Amin = a_H0[gtid][i] - a_tot_D[who][i];
        double Aplu = a_H0[gtid][i] + a_tot_D[who][i];
        double Jmin = h1 * (a_H0[who1][i] - a1_tot_D[who][i]);
        double Jplu = h1 * (a_H0[who1][i] + a1_tot_D[who][i]);
        double Smin = h1 * h1 * (a_H0[who2][i] - a2_tot_D[who][i]);
        double Splu = h1 * h1 * (a_H0[who2][i] + a2_tot_D[who][i]);

        pos_CH[who][i] += h1 * vel_CH[who][i] - 0.4 * h2 * Amin + over * h2 * Jplu;
        vel_CH[who][i] += h1 * Aplu - 0.4 * h1 * Jmin + over * h1 * Splu;
        pos_CH[who][i] += h1 * vel_CH[who][i];

        a3_H[who][i] = h3 * (-5.0 * Amin + 5.0 * Jplu - Smin);
        double a4half = h4 * (-Jmin + Splu);
        double a5half = h5 * (3.0 * Amin - 3.0 * Jplu + Smin);
        a3_H[who][i] += h1 * a4half + 0.5 * h2 * a5half;
        a4half += h1 * a5half;

        if (i == 0) {
            a3_H[who].w = sqrt(a3_H[who][0] * a3_H[who][0] + a3_H[who][1] * a3_H[who][1] + a3_H[who][2] * a3_H[who][2]);
            double a4mod = sqrt(a4half * a4half + a4half * a4half + a4half * a4half);
            double a5mod = sqrt(a5half * a5half + a5half * a5half + a5half * a5half);

            double dt6 = (sqrt(a_H0[gtid].w * a_H0[who2].w) + a_H0[who1].w) / (a5mod * a3_H[who].w + a4mod * a4mod);
            dt6 = ETA6 * pow(dt6, 1.0 / 6.0);

            double stp = h;
            double overh3 = 1.0 / (stp * stp * stp);
            double overh2 = 1.0 / (stp * stp);

            double a2d[3], a3d[3];
            for (int j = 0; j < 3; j++) {
                a2d[j] = overh2 * (-6.0 * (a_tot_D[who][j] - a_H0[gtid][j]) - stp * (4.0 * a_H0[who1][j] + 2.0 * a1_tot_D[who][j]));
                a3d[j] = overh3 * (12.0 * (a_tot_D[who][j] - a_H0[gtid][j]) + 6.0 * stp * (a_H0[who1][j] + a1_tot_D[who][j]));
                a2d[j] += h * a3d[j];
            }

            a_H0[who2].w = a2d[0] * a2d[0] + a2d[1] * a2d[1] + a2d[2] * a2d[2];
            a3_H[who].w = a3d[0] * a3d[0] + a3d[1] * a3d[1] + a3d[2] * a3d[2];

            double dt4 = sqrt(ETA4 * (sqrt(a_H0[gtid].w * a_H0[who2].w) + a_H0[who1].w) / (sqrt(a_H0[who1].w * a3_H[who].w) + a_H0[who2].w));

            double dt = 0.5 * dt4 + 0.5 * dt6;

            double rest = GTIME / (2.0 * step[who]);
            rest = (double)((int)(rest)) - rest;

            if (dt > 2.0 * step[who] && rest == 0.0 && 2.0 * step[who] <= DTMAX)
                step[who] *= 2.0;
            else if (dt < 0.5 * step[who])
                step[who] *= 0.25;
            else if (dt < step[who])
                step[who] *= 0.5;

            if (step[who] < DTMIN)
                step[who] = DTMIN;

            a_tot_D[who] = a_H0[gtid];
            a1_tot_D[who] = a_H0[who1];
            a2_tot_D[who] = a_H0[who2];
        }
    }
}