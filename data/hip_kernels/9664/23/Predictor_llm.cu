#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Predictor (const double TIME, double4 *p_pred, float4  *v_pred, float4  *a_pred, double4 *p_corr, double4 *v_corr, double  *loc_time, double4 *acc, double4 *acc1, double4 *acc2, double4 *acc3, int istart, int* nvec, int ppgpus, unsigned int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x + istart;
    int cost = ppgpus + istart;

    if (i >= cost) {
        i = nvec[i - cost];
        if (i >= istart && i < cost)
            i = -1;
    }
    if (i < 0)
        return;

    double timestep = TIME - loc_time[i];
    double t2 = timestep * timestep * 0.5;
    double t3 = t2 * timestep / 3.0;
    double t4 = t3 * timestep / 4.0;
    double t5 = t4 * timestep / 5.0;

    // Load data from global memory once
    double4 mypcorr = p_corr[i];
    double4 myvcorr = v_corr[i];
    double4 myacc = acc[i];
    double4 myacc1 = acc1[i];
    double4 myacc2 = acc2[i];
    double4 myacc3 = acc3[i];

    // Calculate position prediction
    p_pred[i] = make_double4(
        mypcorr.x + timestep * myvcorr.x + t2 * myacc.x + t3 * myacc1.x + t4 * myacc2.x + t5 * myacc3.x,
        mypcorr.y + timestep * myvcorr.y + t2 * myacc.y + t3 * myacc1.y + t4 * myacc2.y + t5 * myacc3.y,
        mypcorr.z + timestep * myvcorr.z + t2 * myacc.z + t3 * myacc1.z + t4 * myacc2.z + t5 * myacc3.z,
        0.0 // Assuming the w component is not used
    );

    // Calculate velocity prediction
    v_pred[i] = make_float4(
        myvcorr.x + timestep * myacc.x + t2 * myacc1.x + t3 * myacc2.x + t4 * myacc3.x,
        myvcorr.y + timestep * myacc.y + t2 * myacc1.y + t3 * myacc2.y + t4 * myacc3.y,
        myvcorr.z + timestep * myacc.z + t2 * myacc1.z + t3 * myacc2.z + t4 * myacc3.z,
        0.0f // Assuming the w component is not used
    );

    // Calculate acceleration prediction
    a_pred[i] = make_float4(
        myacc.x + timestep * myacc1.x + t2 * myacc2.x + t3 * myacc3.x,
        myacc.y + timestep * myacc1.y + t2 * myacc2.y + t3 * myacc3.y,
        myacc.z + timestep * myacc1.z + t2 * myacc2.z + t3 * myacc3.z,
        0.0f // Assuming the w component is not used
    );
}