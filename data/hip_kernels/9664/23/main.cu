#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <hiprand_kernel.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "Predictor.cu"
#include<chrono>
#include<iostream>
using namespace std;
using namespace std::chrono;
int blocks_[20][2] = {{8,8},{16,16},{24,24},{32,32},{1,64},{1,128},{1,192},{1,256},{1,320},{1,384},{1,448},{1,512},{1,576},{1,640},{1,704},{1,768},{1,832},{1,896},{1,960},{1,1024}};
int matrices_[7][2] = {{240,240},{496,496},{784,784},{1016,1016},{1232,1232},{1680,1680},{2024,2024}};
int main(int argc, char **argv) {
hipSetDevice(0);
char* p;int matrix_len=strtol(argv[1], &p, 10);
for(int matrix_looper=0;matrix_looper<matrix_len;matrix_looper++){
for(int block_looper=0;block_looper<20;block_looper++){
int XSIZE=matrices_[matrix_looper][0],YSIZE=matrices_[matrix_looper][1],BLOCKX=blocks_[block_looper][0],BLOCKY=blocks_[block_looper][1];
const double TIME = 1;
double4 *p_pred = NULL;
hipMalloc(&p_pred, XSIZE*YSIZE);
float4 *v_pred = NULL;
hipMalloc(&v_pred, XSIZE*YSIZE);
float4 *a_pred = NULL;
hipMalloc(&a_pred, XSIZE*YSIZE);
double4 *p_corr = NULL;
hipMalloc(&p_corr, XSIZE*YSIZE);
double4 *v_corr = NULL;
hipMalloc(&v_corr, XSIZE*YSIZE);
double *loc_time = NULL;
hipMalloc(&loc_time, XSIZE*YSIZE);
double4 *acc = NULL;
hipMalloc(&acc, XSIZE*YSIZE);
double4 *acc1 = NULL;
hipMalloc(&acc1, XSIZE*YSIZE);
double4 *acc2 = NULL;
hipMalloc(&acc2, XSIZE*YSIZE);
double4 *acc3 = NULL;
hipMalloc(&acc3, XSIZE*YSIZE);
int istart = 1;
int *nvec = NULL;
hipMalloc(&nvec, XSIZE*YSIZE);
int ppgpus = 1;
unsigned int N = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0)
{
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0)
{
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
Predictor<<<gridBlock,threadBlock>>>(TIME,p_pred,v_pred,a_pred,p_corr,v_corr,loc_time,acc,acc1,acc2,acc3,istart,nvec,ppgpus,N);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 10; ++loop_counter) {
Predictor<<<gridBlock,threadBlock>>>(TIME,p_pred,v_pred,a_pred,p_corr,v_corr,loc_time,acc,acc1,acc2,acc3,istart,nvec,ppgpus,N);
}
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 1000; loop_counter++) {
Predictor<<<gridBlock,threadBlock>>>(TIME,p_pred,v_pred,a_pred,p_corr,v_corr,loc_time,acc,acc1,acc2,acc3,istart,nvec,ppgpus,N);
}
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period> >(end - start);
cout <<'['<<usecs.count()<<','<<'('<<BLOCKX<<','<<BLOCKY<<')' << ','<<'('<<XSIZE<<','<<YSIZE<<')'<<']' << endl;
}
}}