#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dev_get_potential_at_point(float eps2, float *eps, float *xh, float *yh, float *zh, float *xt, float *yt, float *zt, float *phi, int n, float *field_m, float *fxh, float *fyh, float *fzh, float *fxt, float *fyt, float *fzt, int n_field) {
    // Calculate unique thread index for each element
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Iterate while within bounds using a single for-loop
    while (tid < n) {
        float dx, dy, dz, dr2, r, eps2_total;
        // Precompute squared softening length for current particle
        eps2_total = eps2 + eps[tid] * eps[tid];
        // Initialize phi for current thread
        phi[tid] = 0;
        
        // Iterate over field particles
        for (int i = 0; i < n_field; i++) {
            dx = (fxh[i] - xh[tid]) + (fxt[i] - xt[tid]);
            dy = (fyh[i] - yh[tid]) + (fyt[i] - yt[tid]);
            dz = (fzh[i] - zh[tid]) + (fzt[i] - zt[tid]);
            dr2 = dx * dx + dy * dy + dz * dz;
            
            // Check distance is not zero and compute potential
            if (dr2 > 0) {
                r = sqrtf(eps2_total + dr2); // Use sqrtf for better performance
                phi[tid] -= field_m[i] / r;
            }
        }
        
        // Move onto the next set of threads
        tid += blockDim.x * gridDim.x;
    }
}