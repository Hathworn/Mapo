#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reposition(double4 *ac, double4 *ac1, double4 *ac2, double4 *af, unsigned long nextsize)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure valid index and perform coalesced memory writes
    if (i < nextsize) {
        double4 data_ac = ac[i];
        double4 data_ac1 = ac1[i];
        double4 data_ac2 = ac2[i];

        af[i] = data_ac;
        af[i + nextsize] = data_ac1;
        af[i + 2 * nextsize] = data_ac2;
    }
}