#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce(double4 *ac, double4 *ac1, double4 *ac2, unsigned int bf_real, unsigned int dimension) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = dimension * bf_real;
    double4 myacc;

    extern __shared__ double4 shaccelerations[];

    // Combine similar operations using a pointer array for the inputs and outputs
    double4 *input_ac[] = {ac, ac1, ac2};
    double4 *output_ac[] = {ac, ac1, ac2};

    // Determine the data segment for this thread
    int segment = i / k;
    int local_index = i % k;

    if(segment < 3 && local_index < k) {
        myacc = input_ac[segment][local_index];
        shaccelerations[threadIdx.x] = input_ac[segment][local_index + k * segment];

        // Unroll the accumulation loop for efficiency
        myacc.x += shaccelerations[threadIdx.x].x;
        myacc.y += shaccelerations[threadIdx.x].y;
        myacc.z += shaccelerations[threadIdx.x].z;

        output_ac[segment][local_index] = myacc;
    }
}