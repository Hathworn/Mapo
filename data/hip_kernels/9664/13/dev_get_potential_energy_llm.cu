#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dev_get_potential_energy( float *partial_results, float eps2, float *field_m, float *fxh, float *fyh, float *fzh, float *fxt, float *fyt, float *fzt, int n_field) {
    extern __shared__ float thread_results[];
    unsigned int i, j;
    float dx, dy, dz, dr2;
    float potential_energy = 0.0f;

    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int step = blockDim.x * gridDim.x;

    // Coalesced memory access and loop unrolling
    for (j = idx; j < n_field; j += step) {
        float fxhj = fxh[j], fyhj = fyh[j], fzhj = fzh[j];
        float fxtj = fxt[j], fytj = fyt[j], fztj = fzt[j];

        #pragma unroll 4
        for (i = 0; i < j; ++i) {
            dx = (fxh[i] - fxhj) + (fxt[i] - fxtj);
            dy = (fyh[i] - fyhj) + (fyt[i] - fytj);
            dz = (fzh[i] - fzhj) + (fzt[i] - fztj);
            dr2 = dx * dx + dy * dy + dz * dz;
            float r = sqrt(eps2 + dr2);
            potential_energy -= field_m[i] * field_m[j] / r;
        }
    }

    // Reduction for potential energy within a block
    thread_results[threadIdx.x] = potential_energy;
    __syncthreads();
    for (i = blockDim.x / 2; i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            thread_results[threadIdx.x] += thread_results[threadIdx.x + i];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) {
        partial_results[blockIdx.x] = thread_results[0];
    }
}