#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized to run divide operation in parallel by allocating one thread per task
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global thread index
    if (idx == 0) { // Only the first thread performs the divide once
        *accuracy /= N;
    }
}