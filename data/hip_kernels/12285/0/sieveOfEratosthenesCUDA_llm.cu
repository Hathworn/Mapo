#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void sieveOfEratosthenesCUDA(char* primes, uint64_t max) {

    // First thread handles multiples of two
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        primes[0] = 1; // 0 is not prime
        primes[1] = 1; // 1 is not prime
        
        // Mark multiples of two as non-prime
        for (int j = 4; j < max; j += 2) {
            primes[j] = 1;
        }
    } else {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        const uint64_t maxRoot = sqrt((double)max);
        
        // Avoid out of bounds, and skip index 1
        if (index <= maxRoot && index > 1 && primes[index] == 0) {
            // Mark off composite numbers
            for (int j = index * index; j < max; j += index) {
                primes[j] = 1;
            }
        }
    }
}