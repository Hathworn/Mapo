#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void printMat( const double *A, int size )
{
    // Use a single condition to reduce divergence
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        for (int i = 0; i < size; i++) {
            printf("A[%d] = %f\n", i, A[i]);
        }
    }
} /* end printMat */