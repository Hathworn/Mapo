#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void printMat( const double *A, int size )
{
    // Use only one thread to reduce unnecessary launch overhead
    if( blockIdx.x == 0 && threadIdx.x == 0 )
    {
        for( int i = 0; i < size; i++ )
        {
            printf("A[%d] = %f\n", i, A[i]);
        }
    }
} /* end printMat */