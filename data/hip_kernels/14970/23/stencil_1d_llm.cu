#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_1d(int n, double *in, double *out)
{
    // Allocate shared memory (optimize size using dynamic allocation if known at compile-time)
    __shared__ double temp[THREADS_PER_BLOCK + 2 * RADIUS];

    // Calculate global and local indices
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int localIndex = threadIdx.x + RADIUS;

    // Return if the global index is out of bounds
    if (globalIndex >= n) return;

    // Load center element and halos into shared memory
    temp[localIndex] = in[globalIndex];
    if (threadIdx.x < RADIUS) {
        if (globalIndex >= RADIUS)
            temp[localIndex - RADIUS] = in[globalIndex - RADIUS];
        if (globalIndex < n - THREADS_PER_BLOCK)
            temp[localIndex + THREADS_PER_BLOCK] = in[globalIndex + THREADS_PER_BLOCK];
    }

    // Sync threads to ensure all loads are complete
    __syncthreads();

    // Handle boundary conditions
    if (globalIndex < RADIUS || globalIndex >= (n - RADIUS)) {
        out[globalIndex] = (double) globalIndex * (2 * RADIUS + 1.0);
        return;
    }

    // Compute result using loaded shared memory
    double result = 0.0;
    for (int i = -RADIUS; i <= RADIUS; i++) {
        result += temp[localIndex + i];
    }

    // Store the result
    out[globalIndex] = result;
}