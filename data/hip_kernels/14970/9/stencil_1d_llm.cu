#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d(int n, double *in, double *out)
{
    // Allocate shared memory with extra space for halos
    __shared__ double temp[THREADS_PER_BLOCK + 2 * (RADIUS)];

    // Calculate global and local indices
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int localIndex = threadIdx.x + RADIUS;

    // Return if global index exceeds array size
    if (globalIndex >= n) return;

    // Load input elements into shared memory
    temp[localIndex] = in[globalIndex];

    // Load halo elements when within valid bounds
    if (threadIdx.x < RADIUS) {
        if (globalIndex >= RADIUS)
            temp[localIndex - RADIUS] = in[globalIndex - RADIUS];
        if (globalIndex + THREADS_PER_BLOCK < n)
            temp[localIndex + THREADS_PER_BLOCK] = in[globalIndex + THREADS_PER_BLOCK];
    }
    __syncthreads();

    // Handle boundary conditions
    if (globalIndex < RADIUS || globalIndex >= (n - RADIUS)) {
        out[globalIndex] = (double)globalIndex * ((double)RADIUS * 2 + 1);
        return;
    }

    // Calculate result using a stencil operation
    double result = 0.0;
    #pragma unroll  // Unroll stencil loop for performance
    for (int i = -RADIUS; i <= RADIUS; i++) {
        result += temp[localIndex + i];
    }

    out[globalIndex] = result;
}