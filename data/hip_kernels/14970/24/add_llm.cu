#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void add(int *a, int *b, int *c)
{
    // Using block and thread IDs for scalability
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    c[idx] = a[idx] + b[idx];
}
```
