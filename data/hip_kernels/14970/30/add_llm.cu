#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Get global index of each thread using built-in function for performance
    int index = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x; 
    // Perform addition and store the result
    c[index] = a[index] + b[index];
}