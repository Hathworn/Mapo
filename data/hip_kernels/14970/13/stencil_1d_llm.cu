#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_1d(int n, double *in, double *out)
{
    // Calculate global index in the array
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Return if my global index is larger than the array size
    if (globalIndex >= n) return;

    // Handle boundary conditions
    if (globalIndex < RADIUS || globalIndex >= (n - RADIUS))
    {
        out[globalIndex] = (double) globalIndex * ((double)RADIUS * 2 + 1);
        return;
    }

    double result = 0.0;

    // Use shared memory to optimize memory accesses
    extern __shared__ double shared_in[];
    int localIndex = threadIdx.x + RADIUS;
    shared_in[localIndex] = in[globalIndex];

    // Load halo elements
    if (threadIdx.x < RADIUS)
    {
        shared_in[localIndex - RADIUS] = in[globalIndex - RADIUS];
        shared_in[localIndex + blockDim.x] = in[globalIndex + blockDim.x];
    }

    __syncthreads();

    // Calculate stencil result
    for (int i = -RADIUS; i <= RADIUS; i++)
    {
        result += shared_in[localIndex + i];
    }

    out[globalIndex] = result;
}