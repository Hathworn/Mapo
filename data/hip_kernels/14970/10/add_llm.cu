#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Use thread index for parallel addition
    int idx = threadIdx.x;
    c[idx] = a[idx] + b[idx];
}