#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c)
{
    // Calculate global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Add vectors
    c[idx] = a[idx] + b[idx];
}