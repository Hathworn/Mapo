#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Using thread index to allow for processing more elements if needed
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Assuming the number of threads handle processing each element
    c[index] = a[index] + b[index];
}