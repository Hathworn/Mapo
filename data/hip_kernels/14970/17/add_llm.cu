#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c)
{
    // Calculate index with grid-stride loop
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = index; i < N; i += stride) {
        c[i] = a[i] + b[i];
    }
}