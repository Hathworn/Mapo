#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel to avoid using printf
__global__ void mykernel() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate unique thread index
    if (idx == 0) {  // Use single thread to output message
        printf("Hello world from device!\n");
    }
} /* end kernel */