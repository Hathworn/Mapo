#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mykernel(){
    // Optimize by reducing I/O operations inside the kernel
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    if (threadId == 0) { // Print once from a single thread
        printf("Hello world from device!\n");
    }
} /* end kernel */