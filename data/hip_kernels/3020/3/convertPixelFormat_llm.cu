#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertPixelFormat(uint8_t* inputBgra, uint8_t* outputYuv, int numPixels) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    
    // Optimize by unrolling the conversion loop for better performance
    for (; idx < numPixels; idx += stride) {
        short3 yuv16;
        char3 yuv8;

        // Load input only once per pixel and store in registers
        uint8_t b = inputBgra[idx * 4];
        uint8_t g = inputBgra[idx * 4 + 1];
        uint8_t r = inputBgra[idx * 4 + 2];

        // Efficient YUV conversion
        yuv16.x = 66 * r + 129 * g + 25 * b;
        yuv16.y = -38 * r - 74 * g + 112 * b;
        yuv16.z = 112 * r - 94 * g - 18 * b;

        // Normalize and offset YUV values
        yuv8.x = (yuv16.x >> 8) + 16;
        yuv8.y = (yuv16.y >> 8) + 128;
        yuv8.z = (yuv16.z >> 8) + 128;

        // Output result
        *(reinterpret_cast<char3*>(&outputYuv[idx * 3])) = yuv8;
    }
}