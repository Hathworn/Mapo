#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_unpack_yuy2_rgb8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Check bounds only once outside the loop
    if (i >= superPixCount) return;

    int stride = blockDim.x * gridDim.x;

    // Use loop unrolling to process two super pixels per iteration
    #pragma unroll 2
    for (; i < superPixCount; i += stride) {
        int idx = i * 4;

        uint8_t y0 = src[idx];
        uint8_t u0 = src[idx + 1];
        uint8_t y1 = src[idx + 2];
        uint8_t v0 = src[idx + 3];

        int16_t c = y0 - 16;
        int16_t d = u0 - 128;
        int16_t e = v0 - 128;

        int32_t t;
        #define clamp(x)  ((t = (x)) > 255 ? 255 : t < 0 ? 0 : t)

        int odx = i * 6;

        // Precompute common term
        int32_t c298 = 298 * c;

        dst[odx] = clamp((c298 + 409 * e + 128) >> 8);
        dst[odx + 1] = clamp((c298 - 100 * d - 409 * e + 128) >> 8);
        dst[odx + 2] = clamp((c298 + 516 * d + 128) >> 8);

        c = y1 - 16;
        c298 = 298 * c; // Reuse c298 variable

        dst[odx + 3] = clamp((c298 + 409 * e + 128) >> 8);
        dst[odx + 4] = clamp((c298 - 100 * d - 409 * e + 128) >> 8);
        dst[odx + 5] = clamp((c298 + 516 * d + 128) >> 8);

        #undef clamp
    }
}