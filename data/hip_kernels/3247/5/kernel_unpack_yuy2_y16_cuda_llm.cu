#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_unpack_yuy2_y16_cuda(const uint8_t *src, uint8_t *dst, int superPixCount)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Unroll loop by processing multiple elements per thread to utilize memory coalescing
    int idx = i * 4; // Pre-calculate index for better performance
    if (i < superPixCount)
    {
        dst[idx] = 0;
        dst[idx + 1] = src[idx + 0];
        dst[idx + 2] = 0;
        dst[idx + 3] = src[idx + 2];
    }
}