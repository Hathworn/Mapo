#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ uint8_t clamp(int32_t x) {
    return x > 255 ? 255 : x < 0 ? 0 : x;
}

__global__ void kernel_unpack_yuy2_rgba8_cuda(const uint8_t *src, uint8_t *dst, int superPixCount) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (i < superPixCount) {
        int maxIndex = (superPixCount + stride - 1) / stride * stride;
        
        for (; i < maxIndex; i += stride) {
            if (i >= superPixCount) break;  // Prevent out-of-bound access

            int idx = i * 4;
            uint8_t y0 = src[idx];
            uint8_t u0 = src[idx + 1];
            uint8_t y1 = src[idx + 2];
            uint8_t v0 = src[idx + 3];

            int16_t c = y0 - 16;
            int16_t d = u0 - 128;
            int16_t e = v0 - 128;

            int odx = i * 8;

            dst[odx] = clamp((298 * c + 409 * e + 128) >> 8);
            dst[odx + 1] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
            dst[odx + 2] = clamp((298 * c + 516 * d + 128) >> 8);
            dst[odx + 3] = 255;

            c = y1 - 16;

            dst[odx + 4] = clamp((298 * c + 409 * e + 128) >> 8);
            dst[odx + 5] = clamp((298 * c - 100 * d - 409 * e + 128) >> 8);
            dst[odx + 6] = clamp((298 * c + 516 * d + 128) >> 8);
            dst[odx + 7] = 255;
        }
    }
}