#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_unpack_yuy2_bgr8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int t;

    // Moved calculations outside of the loop for better performance.
    for (; i < superPixCount; i += stride) 
    {
        int idx = i * 4;

        uint8_t y0 = src[idx];
        uint8_t u0 = src[idx + 1];
        uint8_t y1 = src[idx + 2];
        uint8_t v0 = src[idx + 3];

        int16_t c0 = y0 - 16;
        int16_t c1 = y1 - 16;
        int16_t d = u0 - 128;
        int16_t e = v0 - 128;

        int odx = i * 6;

        // Precompute common expressions
        int32_t d_c = 516 * d + 128;
        int32_t e_c = 409 * e + 128;
        int32_t de_c = -100 * d - 409 * e + 128;

        #define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

        dst[odx + 2] = clamp((298 * c0 + e_c) >> 8);
        dst[odx + 1] = clamp((298 * c0 + de_c) >> 8);
        dst[odx] = clamp((298 * c0 + d_c) >> 8);

        dst[odx + 5] = clamp((298 * c1 + e_c) >> 8);
        dst[odx + 4] = clamp((298 * c1 + de_c) >> 8);
        dst[odx + 3] = clamp((298 * c1 + d_c) >> 8);

        #undef clamp
    }
}