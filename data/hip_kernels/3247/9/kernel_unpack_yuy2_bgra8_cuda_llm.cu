#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_unpack_yuy2_bgra8_cuda(const uint8_t * src, uint8_t *dst, int superPixCount)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    if (i >= superPixCount)
        return;

    // Optimize loop condition and avoid recalculating stride and thread index 
    int idx = i * 4;
    int odx = i * 8;

    for (; i < superPixCount; i += stride, idx += stride * 4, odx += stride * 8) {

        uint8_t y0 = src[idx];
        uint8_t u0 = src[idx + 1];
        uint8_t y1 = src[idx + 2];
        uint8_t v0 = src[idx + 3];

        int16_t c0 = y0 - 16;
        int16_t d = u0 - 128;
        int16_t e = v0 - 128;
        int32_t t;

        #define clamp(x)  ((t=(x)) > 255 ? 255 : t < 0 ? 0 : t)

        dst[odx + 3] = 255;
        dst[odx + 2] = clamp((298 * c0 + 409 * e + 128) >> 8);
        dst[odx + 1] = clamp((298 * c0 - 100 * d - 409 * e + 128) >> 8);
        dst[odx] = clamp((298 * c0 + 516 * d + 128) >> 8);

        int16_t c1 = y1 - 16;

        dst[odx + 7] = 255;
        dst[odx + 6] = clamp((298 * c1 + 409 * e + 128) >> 8);
        dst[odx + 5] = clamp((298 * c1 - 100 * d - 409 * e + 128) >> 8);
        dst[odx + 4] = clamp((298 * c1 + 516 * d + 128) >> 8);

        #undef clamp
    }
}