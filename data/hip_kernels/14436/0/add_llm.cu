#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NUM 100

__global__ void add (int *a, int *b, int *c)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (idx < NUM) { // Ensure index is within bounds
        c[idx] = a[idx] + b[idx]; // Perform addition for each element
    }
}