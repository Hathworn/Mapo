#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

int test_reduce(int* v);

using namespace std;

__global__ void reduce0(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];
    // Load elements from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < blockDim.x * gridDim.x) { // Check bounds to prevent out-of-bounds memory access
        sdata[tid] = g_idata[i];
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();

    // Reduction in shared memory
    for(unsigned int s = blockDim.x / 2; s > 0; s >>= 1) { // Start from half block size and use bit shift
        if (tid < s) {// Ensure threads operate within bounds
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}