#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Using single index calculation for thread ID to improve memory access coalescing and avoid branch divergence.
    int idx = blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * w;

    if (idx < w * h)
    {
        image[idx] = value;
    }
}