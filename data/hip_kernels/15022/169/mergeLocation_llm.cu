#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate overall thread index
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within bounds
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];
        // Use registers to store scaled x and y values
        float scaledX = loc.x * scale;
        float scaledY = loc.y * scale;

        // Assign scaled values to the output arrays
        x[ptidx] = scaledX;
        y[ptidx] = scaledY;
    }
}