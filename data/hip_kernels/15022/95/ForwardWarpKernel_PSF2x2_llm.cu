#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // Get fractional parts and integer pixel coordinates
    float px, py, dx = modff(cx, &px), dy = modff(cy, &py);
    int tx = (int)px, ty = (int)py;
    float value = src[image_row_offset + j];
    float weight;

    // Use shared memory for tile-specific weight accumulation
    __shared__ float shmem_dst[32][32]; // Adjust size based on max block size
    __shared__ float shmem_norm[32][32]; // Adjust size as needed
    shmem_dst[threadIdx.y][threadIdx.x] = 0.0f;
    shmem_norm[threadIdx.y][threadIdx.x] = 0.0f;
    __syncthreads();

    // Process target pixel and contribute to shared memory
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0))) {
        weight = dx * dy;
        atomicAdd(&shmem_dst[threadIdx.y][threadIdx.x], value * weight);
        atomicAdd(&shmem_norm[threadIdx.y][threadIdx.x], weight);
    }

    // Bottom left pixel
    tx -= 1;
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0))) {
        weight = (1.0f - dx) * dy;
        atomicAdd(&shmem_dst[threadIdx.y][threadIdx.x], value * weight);
        atomicAdd(&shmem_norm[threadIdx.y][threadIdx.x], weight);
    }

    // Upper left pixel
    ty -= 1;
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0))) {
        weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(&shmem_dst[threadIdx.y][threadIdx.x], value * weight);
        atomicAdd(&shmem_norm[threadIdx.y][threadIdx.x], weight);
    }

    // Upper right pixel
    tx += 1;
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0))) {
        weight = dx * (1.0f - dy);
        atomicAdd(&shmem_dst[threadIdx.y][threadIdx.x], value * weight);
        atomicAdd(&shmem_norm[threadIdx.y][threadIdx.x], weight);
    }
    
    __syncthreads();

    // Copy shared memory results to global memory
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0))) {
        atomicAdd(dst + ty * image_stride + tx, shmem_dst[threadIdx.y][threadIdx.x]);
        atomicAdd(normalization_factor + ty * image_stride + tx, shmem_norm[threadIdx.y][threadIdx.x]);
    }
}