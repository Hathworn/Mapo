#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Use blockIdx.y and threadIdx.y for i
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Use blockIdx.x and threadIdx.x for j

    if (i < h && j < w) // Combine bounds check into one conditional
    {
        const int pos = i * s + j;
        float scale = normalization_factor[pos];

        // Use ternary and fused multiply-add to improve performance
        image[pos] *= (scale == 0.0f) ? 1.0f : __fdividef(1.0f, scale); 
    }
}