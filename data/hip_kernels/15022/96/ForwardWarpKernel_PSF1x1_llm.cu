#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

// Kernel with optimized access and operations
__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_idx = i * flow_stride + j;  // Compute the flow index once
    int image_idx = i * image_stride + j;  // Compute the image index once

    float u_ = u[flow_idx];
    float v_ = v[flow_idx];

    float cx = u_ * time_scale + (float)j + 1.0f;
    float cy = v_ * time_scale + (float)i + 1.0f;

    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    float value = src[image_idx];  // Load source value

    // Use atomicAdd for valid pixel contribution
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        _atomicAdd(dst + ty * image_stride + tx, value);
    }
}