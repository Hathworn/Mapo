#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void finish(unsigned int* counter) {
    __syncthreads();
    __threadfence();
    if (threadIdx.x == 0) { 
        atomicAdd(counter, 1); 
    }
}

__global__ void GRUPrepare(unsigned int* finished, const int round) {
    // Calculate global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use all threads to initialize `finished` array if large; handle out of bounds
    for (int i = idx; i < round; i += gridDim.x * blockDim.x) {
        finished[i] = 0;
    }
}