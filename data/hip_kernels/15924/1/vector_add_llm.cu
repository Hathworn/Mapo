#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function for vector addition
__global__ void vector_add(double const *A, double const *B, double *C, int const N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we don't write to C when i is out of bounds
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}