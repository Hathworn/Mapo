#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(int *dest, int a, int b)
{
    // Use shared memory to store the sum temporarily
    __shared__ int temp_sum;
    
    // Each thread in the block calculates the sum
    // Assuming only one thread is used; no reduction is needed
    if (threadIdx.x == 0) {
        temp_sum = a + b;
    }

    // Use __syncthreads() to ensure all threads have computed before copying to global memory
    __syncthreads();

    // Copy from shared to global memory
    if (threadIdx.x == 0) {
        *dest = temp_sum;
    }
}