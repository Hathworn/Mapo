#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void myTextureKernel(hipSurfaceObject_t SurfObj, size_t width, size_t height)
{
    // Calculate 2D thread indices
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds for thread execution
    if (idx < width && idy < height) {
        uchar4 data = make_uchar4(255, 255, 255, 255);
        // Write to output surface
        surf2Dwrite(data, SurfObj, idx * sizeof(uchar4), idy);
    }
}