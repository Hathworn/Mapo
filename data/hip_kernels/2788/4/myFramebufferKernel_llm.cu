#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to calculate pixel color in parallel
__global__ void myFramebufferKernelOptimized(int width, int height, float *framebuffer)
{
    // Calculate the pixel indices
    int x = blockIdx.x * blockDim.x + threadIdx.x; // x-coordinate of pixel
    int y = blockIdx.y * blockDim.y + threadIdx.y; // y-coordinate of pixel

    // Ensure the thread corresponds to a valid pixel position
    if (x < width && y < height)
    {
        // Calculate linear pixel index
        int index = y * width + x;

        // Set color using some sample calculation, can be replaced with actual logic
        framebuffer[index * 3 + 0] = 0.5f; // Red component
        framebuffer[index * 3 + 1] = 0.5f; // Green component
        framebuffer[index * 3 + 2] = 0.5f; // Blue component
    }
}
