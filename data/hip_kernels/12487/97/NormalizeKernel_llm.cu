#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate global row and col indices
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < h && j < w) {  // Simplify boundary check
        const int pos = i * s + j;
        
        float scale = normalization_factor[pos];
        
        float invScale = (scale == 0.0f) ? 1.0f : __frcp_rn(scale); // Use fast reciprocal
        
        image[pos] *= invScale;
    }
}