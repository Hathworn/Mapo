#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Use shared memory for dst buffer to reduce global memory access
    extern __shared__ float s_dst[];

    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // bottom left corner of target pixel
    float cx = u_ * time_scale + (float)j + 1.0f;
    float cy = v_ * time_scale + (float)i + 1.0f;
    // pixel containing bottom left corner
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    float value = src[image_row_offset + j];
    // fill pixel
    if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0)))
    {
        // Use atomic operations in shared memory instead of global memory directly
        atomicAdd(s_dst + ty * image_stride + tx, value);
    }
    // Synchronize shared memory changes
    __syncthreads();

    // Write results back to global memory
    tx = threadIdx.x + blockDim.x * blockIdx.x;
    ty = threadIdx.y + blockDim.y * blockIdx.y;
    if (ty < h && tx < w)
    {
        dst[ty * image_stride + tx] = s_dst[ty * image_stride + tx];
    }
}