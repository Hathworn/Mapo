#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate unique thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x + 
                (blockIdx.y * blockDim.y + threadIdx.y) * w;

    // Ensure the thread is within bounds
    if (index < w * h) {
        image[index] = value;
    }
}