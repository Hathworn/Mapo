#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by applying loop unrolling to minimize loop overhead
    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;

        // Unrolling may occur when there are more points per thread
        if (ptidx + blockDim.x < npoints)
        {
            loc = loc_[ptidx + blockDim.x];
            x[ptidx + blockDim.x] = loc.x * scale;
            y[ptidx + blockDim.x] = loc.y * scale;
        }
    }
}