#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + static_cast<float>(i) + 1.0f;
    // Pixel containing bottom left corner
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    // Target pixel integer coords
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);
    float value = src[image_row_offset + j];
    float weight;

    // Early exit if target coordinates are out of bounds (for optimization)
    if (tx < 0 || tx >= w || ty < 0 || ty >= h) return;

    // Fill pixel containing bottom right corner
    weight = dx * dy;
    _atomicAdd(dst + ty * image_stride + tx, value * weight);
    _atomicAdd(normalization_factor + ty * image_stride + tx, weight);

    // Fill pixel containing bottom left corner
    if (tx - 1 >= 0)
    {
        weight = (1.0f - dx) * dy;
        _atomicAdd(dst + ty * image_stride + (tx - 1), value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + (tx - 1), weight);
    }

    // Fill pixel containing upper left corner
    if (ty - 1 >= 0)
    {
        weight = (1.0f - dx) * (1.0f - dy);
        _atomicAdd(dst + (ty - 1) * image_stride + (tx - 1), value * weight);
        _atomicAdd(normalization_factor + (ty - 1) * image_stride + (tx - 1), weight);
    }

    // Fill pixel containing upper right corner
    if (ty - 1 >= 0 && tx >= 0)
    {
        weight = dx * (1.0f - dy);
        _atomicAdd(dst + (ty - 1) * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + (ty - 1) * image_stride + tx, weight);
    }
}