#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < N) {
        // Use temporary variable to avoid redundant memory loads
        float a = A[i];
        float b = B[i];
        C[i] = a + b;
    }
}