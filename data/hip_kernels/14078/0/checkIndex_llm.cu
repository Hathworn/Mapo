#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void checkIndex(void) {
    // Compute unique thread index based on block and thread indices
    int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    
    // Optimize printf by minimizing the number of parameters
    printf("Thread ID: %d || Block ID: %d\n", threadId, blockId);
}