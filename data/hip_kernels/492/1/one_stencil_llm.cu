#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/////////////////////////////////////////////////////////

// Computes the 1-stencil using GPUs.

#define BLOCK_SIZE 1024
#define WARP_SIZE 32

#ifndef k
#define k 3
#endif
#ifndef OUTPUT_PER_THREAD
#define OUTPUT_PER_THREAD 1
#endif
#define LOCAL_REGISTER_SIZE ((1+OUTPUT_PER_THREAD) > (k+31)/32 ? (1+OUTPUT_PER_THREAD) : (k+31)/32)
#ifndef TEST_TIMES
#define TEST_TIMES 5
#endif

float host_k_stencil (int *A, int *B, int sizeOfA, int withRc);

__global__ void one_stencil (int *A, int *B, int sizeOfA)
{
    extern __shared__ int s[];
    
    int localId = threadIdx.x;
    int startOfBlock = blockIdx.x * blockDim.x;
    int globalId = localId + startOfBlock;

    if (globalId >= sizeOfA)
        return;

    // Load data into shared memory with boundary check
    s[localId] = A[globalId];
    if (localId < 2 && blockDim.x + globalId < sizeOfA) {
        s[blockDim.x + localId] = A[blockDim.x + globalId];
    }
    
    __syncthreads(); // Synchronize threads for shared memory access

    // Compute result using shared memory
    if (globalId < sizeOfA - 2) {
        B[globalId] = s[localId] + s[localId + 1] + s[localId + 2];
    }
}