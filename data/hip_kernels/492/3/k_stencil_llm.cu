#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/////////////////////////////////////////////////////////

#define BLOCK_SIZE 1024
#define WARP_SIZE 32

#ifndef k
#define k 3
#endif
#ifndef OUTPUT_PER_THREAD
#define OUTPUT_PER_THREAD 1
#endif
#define LOCAL_REGISTER_SIZE ((1+OUTPUT_PER_THREAD) > (k+31)/32 ? (1+OUTPUT_PER_THREAD) : (k+31)/32)
#ifndef TEST_TIMES
#define TEST_TIMES 5
#endif

float host_k_stencil (int *A, int *B, int sizeOfA, int withRc);

__global__ void k_stencil (int *A, int *B, int sizeOfA)
{
    extern __shared__ int s[];
    int localId = threadIdx.x;
    int startOfBlock = blockIdx.x * blockDim.x * OUTPUT_PER_THREAD;
    int globalId = localId + startOfBlock;

    if (globalId >= sizeOfA)
        return;

    // Efficient loading with boundary checks
    for (int i = 0; i < OUTPUT_PER_THREAD && (globalId + i * BLOCK_SIZE) < sizeOfA; ++i)
    {
        s[localId + i * BLOCK_SIZE] = A[globalId + i * BLOCK_SIZE];
    }

    if (localId < k && (globalId + blockDim.x * OUTPUT_PER_THREAD) < sizeOfA)
    {
        s[localId + blockDim.x * OUTPUT_PER_THREAD] = A[globalId + blockDim.x * OUTPUT_PER_THREAD];
    }
    
    __syncthreads();

    // Re-use of computed results for better throughput
    for (int j = 0; j < OUTPUT_PER_THREAD && (globalId + j * BLOCK_SIZE) < (sizeOfA - k); ++j)
    {
        int sum = 0;
        for (int i = 0; i < k + 1; ++i)
        {
            sum += s[localId + j * BLOCK_SIZE + i];
        }
        B[globalId + BLOCK_SIZE * j] = sum;
    }
}