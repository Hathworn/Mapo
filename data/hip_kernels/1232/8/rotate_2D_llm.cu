#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rotate_2D(float* coords, size_t dim_y, size_t dim_x, float cos_angle, float sin_angle){
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = dim_x * dim_y;
    
    if(index < total){
        // Pre-calculate indices
        size_t idx_y = index;
        size_t idx_x = index + total;
        
        // Fetch old coordinates
        float old_y = coords[idx_y];
        float old_x = coords[idx_x];
        
        // Calculate new coordinates
        float new_y = cos_angle * old_y + sin_angle * old_x;
        float new_x = -sin_angle * old_y + cos_angle * old_x;
        
        // Write new coordinates
        coords[idx_y] = new_y;
        coords[idx_x] = new_x;
    }
}