#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ int mirror(int index, int len){
    int s2 = 2 * len - 2;
    if(index < 0){
        index = s2 * (-index / s2) + index;
        return index <= 1 - len ? index + s2 : -index;
    }
    if(index >= len){
        index -= s2 * (index / s2);
        if(index >= len)
            index = s2 - index;
        return index;
    }
    if(index < 0 || index >= len) index = mirror(index, len);
    return index;
}

__global__ void gussain_filter_x(float* random, float* kernel, int lw, size_t dim_z, size_t dim_y, size_t dim_x, int mode, float cval){
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = dim_x * dim_y * dim_z;
    size_t total_xy = dim_x * dim_y;
    size_t id_x = index % dim_x;
    size_t id_y = (index / dim_x) % dim_y;
    size_t id_z = (index / total_xy) % dim_z;
    size_t id_block = index / total;
    
    if(index >= total * (dim_z > 1 ? 3 : 2)) return; // Exit early if index is out of bounds

    float new_pixel = 0.0f;
    
    int start = -lw;
    int end = lw + 1;
    int block_offset = id_block * total + id_z * total_xy + id_y * dim_x; // Precompute block offset

    if(mode == 0){
        for(int i = start; i < end; ++i){
            int id = id_x + i;
            // Clamp with constant value if out of bounds
            new_pixel += ((id < 0 || id >= dim_x) ? cval : kernel[i + lw] * random[block_offset + id]) * kernel[i + lw];
        }
    } else {
        for(int i = start; i < end; ++i){
            int id = mirror(id_x + i, dim_x);
            // Use mirrored index if out of bounds
            new_pixel += kernel[i + lw] * random[block_offset + id];
        }
    }
    __syncthreads();
    random[index] = new_pixel; // Write result to memory
    __syncthreads();
}