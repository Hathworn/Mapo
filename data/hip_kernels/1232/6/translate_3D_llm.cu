#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void translate_3D(float* coords, size_t dim_z, size_t dim_y, size_t dim_x, float seg_z, float seg_y, float seg_x) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = dim_x * dim_y * dim_z;
    
    // Check index is within bounds
    if(index < total) {
        // Apply translations without __syncthreads(); removed as there is no shared memory usage 
        coords[index] += seg_z;
        coords[index + total] += seg_y;
        coords[index + total * 2] += seg_x;
    }
}