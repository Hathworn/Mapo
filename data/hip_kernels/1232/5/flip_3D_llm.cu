#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void exchange(float &a, float &b){
    float temp = a;
    a = b;
    b = temp;
}

__global__ void flip_3D(float* coords, size_t dim_z, size_t dim_y, size_t dim_x, int do_z, int do_y, int do_x){
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = dim_x * dim_y * dim_z;
    size_t total_xy = dim_x * dim_y;
    size_t id_x = index % dim_x;
    size_t id_y = (index / dim_x) % dim_y; // Fixed indexing
    size_t id_z = index / (dim_x * dim_y);

    if(index < total){
        // Remove __syncthreads(); as each branch works on disjoint data
        if(do_x && id_x < (dim_x / 2)){
            exchange(coords[2 * total + id_z * total_xy + id_y * dim_x + id_x],
                     coords[2 * total + id_z * total_xy + id_y * dim_x + dim_x-1 - id_x]);
        }
        if(do_y && id_y < (dim_y / 2)){
            exchange(coords[total + id_z * total_xy + id_y * dim_x + id_x],
                     coords[total + id_z * total_xy + (dim_y-1 - id_y) * dim_x + id_x]);
        }
        if(do_z && id_z < (dim_z / 2)){
            exchange(coords[id_z * total_xy + id_y * dim_x + id_x],
                     coords[(dim_z-1 -id_z) * total_xy + id_y * dim_x + id_x]);
        }
    }
}