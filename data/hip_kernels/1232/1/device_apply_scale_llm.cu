#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_apply_scale(float* coords, float scale, size_t total_size) {
    // Calculate the global index once
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    // Use a loop with a stride for better GPU utilization
    for (; i < total_size; i += stride) {
        coords[i] = coords[i] * scale;
    }
    // No need for __syncthreads(); as there is no shared memory race condition
}