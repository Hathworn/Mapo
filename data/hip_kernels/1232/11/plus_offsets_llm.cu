#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void plus_offsets(float *coords, float *random, size_t total_size, float alpha) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < total_size) {
        // Combined calculations in a single line for clarity and efficiency
        coords[index] += random[index] * alpha;
    }
    // Removed __syncthreads() as it is unnecessary here, for performance
}