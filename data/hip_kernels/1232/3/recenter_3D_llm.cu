#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void recenter_3D(float* coords, size_t dim_z, size_t dim_y, size_t dim_x) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = dim_x * dim_y * dim_z;
    
    // Ensure index is within bounds to prevent unnecessary calculations
    if (index < total) {
        float z_center = (float)dim_z / 2.0f; // Precompute to reduce repeated calculations
        float y_center = (float)dim_y / 2.0f; 
        float x_center = (float)dim_x / 2.0f; 

        coords[index] += z_center;
        coords[index + total] += y_center;
        coords[index + 2 * total] += x_center;
    }
}