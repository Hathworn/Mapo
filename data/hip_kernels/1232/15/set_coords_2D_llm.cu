#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void set_coords_2D(float* coords, size_t y, size_t x) {
    // Calculate global index in flattened grid
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (index < x * y) {
        // Calculate 2D indices from flattened index
        size_t id_x = index % x;
        size_t id_y = index / x;
        
        // Calculate and set coordinates
        coords[index] = id_y - (float)y / 2.0;
        coords[index + x * y] = id_x - (float)x / 2.0;
    }
    // No need for __syncthreads() as there are no shared resources.
}