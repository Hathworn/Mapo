#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void recenter_2D(float* coords, size_t dim_y, size_t dim_x) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total_elements = dim_x * dim_y;
    // Remove __syncthreads as it is not needed; each thread works on unique data.
    if (index < total_elements) {
        float half_dim_y = (float)dim_y / 2.0f; // Pre-compute once per thread.
        float half_dim_x = (float)dim_x / 2.0f; // Pre-compute once per thread.
        coords[index] += half_dim_y;
        coords[index + total_elements] += half_dim_x;
    }
}