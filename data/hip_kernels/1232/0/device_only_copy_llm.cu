#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void device_only_copy(float* output, float* input, size_t total_size) {
    // Calculate global thread index
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop unrolling for efficiency
    if (i < total_size) {
        output[i] = input[i];
    }
    __syncthreads();
}