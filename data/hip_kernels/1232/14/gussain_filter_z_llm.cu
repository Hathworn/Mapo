#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ int mirror(int index, int len) {
    int s2 = 2 * len - 2;
    if (index < 0) {
        index = s2 * (-index / s2) + index;
        return index <= 1 - len ? index + s2 : -index;
    }
    if (index >= len) {
        index -= s2 * (index / s2);
        if (index >= len)
            index = s2 - index;
        return index;
    }
    if (index < 0 || index >= len) index = mirror(index, len);
    return index;
}

__global__ void gussain_filter_z(float* random, float* kernel, int lw, size_t dim_z, size_t dim_y, size_t dim_x, int mode, float cval) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = dim_x * dim_y * dim_z;
    size_t total_xy = dim_x * dim_y;

    // Exit if index is out of the total size
    if (index >= total) return;

    size_t id_x = index % dim_x;
    size_t id_y = (index / dim_x) % dim_y;
    size_t id_z = (index / total_xy) % dim_z;
    size_t id_block = index / total;
    
    float new_pixel = 0;

    // Pre-calculate offset for the base address
    size_t base_offset = id_block * total + id_y * dim_x + id_x;

    if (mode == 0) {
        for (int i = -lw; i <= lw; i++) {
            int id = id_z + i;
            new_pixel += (id < 0 || id >= dim_z) ? cval * kernel[i + lw] : 
                         kernel[i + lw] * random[base_offset + id * total_xy];
        }
    } else {
        for (int i = -lw; i <= lw; i++) {
            int id = id_z + i;
            id = mirror(id, dim_z);
            new_pixel += kernel[i + lw] * random[base_offset + id * total_xy];
        }
    }
    
    // Only update the random array for valid indices
    random[index] = new_pixel;
}