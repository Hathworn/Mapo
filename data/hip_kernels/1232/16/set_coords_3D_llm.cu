#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_coords_3D(float* coords, size_t z, size_t y, size_t x) {
    // Calculate global thread index
    size_t index = blockIdx.x * blockDim.x + threadIdx.x; 

    // Compute 3D coordinates only if index is within bounds
    if (index < x * y * z) {
        size_t xy = x * y; // Cache frequently used value
        size_t id_z = index / xy; 
        size_t id_y = (index % xy) / x; 
        size_t id_x = index % x; 

        // Compute and write coordinates to global memory
        coords[index] = id_z - (float)z / 2.0f;
        coords[index + xy] = id_y - (float)y / 2.0f;
        coords[index + 2 * xy] = id_x - (float)x / 2.0f;
    }
}