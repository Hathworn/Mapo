#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void exchange(float &a, float &b) {
    float temp = a;
    a = b;
    b = temp;
}

__global__ void flip_2D(float* coords, size_t dim_y, size_t dim_x, int do_y, int do_x) {
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = dim_x * dim_y;

    // Parallelize work across x and y dimensions
    if (index < total) {
        size_t id_x = index % dim_x;
        size_t id_y = index / dim_x;

        // Handle flipping in x dimension
        if (do_x && id_x < (dim_x / 2)) {
            exchange(coords[total + id_y * dim_x + id_x],
                     coords[total + id_y * dim_x + dim_x - 1 - id_x]);
        }

        // Handle flipping in y dimension
        if (do_y && id_y < (dim_y / 2)) {
            exchange(coords[id_y * dim_x + id_x],
                     coords[(dim_y - 1 - id_y) * dim_x + id_x]);
        }
    }
}