#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    // Calculate the unique global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if thread index is out of bounds
    if (index >= n) return;

    // Initialize sum variable
    float sum = 0.0f;

    // Unroll loop to optimize memory access
    for(int b = 0; b < batch; ++b) {
        int i = b * n + index;
        sum += delta[i];
    }

    // Update bias with computed sum
    atomicAdd(&bias_updates[index], sum);
}