#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    // Calculate the global thread ID more efficiently
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Compute grid stride

    // Use loop unrolling technique for optimization
    for (int id = index; id < n * size * batch; id += stride) {
        int i = id % size;
        int j = (id / size) % n;
        int k = id / (size * n);
        output[(k * n + j) * size + i] += biases[j];
    }
}