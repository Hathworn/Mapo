#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Use shared memory for mask_num to reduce global memory access
    __shared__ float shared_mask_num;
    if (threadIdx.x == 0) shared_mask_num = mask_num;
    __syncthreads();

    // Use efficient memory access pattern
    if (i < n && mask[i] == shared_mask_num) {
        x[i] *= scale;
    }
}