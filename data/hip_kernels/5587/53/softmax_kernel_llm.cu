#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    extern __shared__ float shared[];  // Use shared memory for efficiency
    float* shared_output = shared;
    float sum = 0;
    float largest = -INFINITY;

    // Collaborative loading of data into shared memory
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        float val = input[i * stride];
        shared_output[i] = val;
        atomicMax(&largest, val); // Determine largest value
    }
    __syncthreads();

    // Compute exponentials and store in shared memory
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        shared_output[i] = expf((shared_output[i] - largest) / temp);
        atomicAdd(&sum, shared_output[i]); // Sum up the exponentials
    }
    __syncthreads();

    // Normalize the output values
    for (int i = threadIdx.x; i < n; i += blockDim.x) {
        output[i * stride] = shared_output[i] / sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= batch * groups) return;

    int b = id / groups;
    int g = id % groups;
    softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}