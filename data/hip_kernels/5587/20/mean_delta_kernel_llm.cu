#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return; // Check bounds
    
    float sum_delta = 0.0f; // Local variable for sum
    
    // Loop unrolling for reduced loop overhead
    for (int j = 0; j < batch; ++j) {
        int batch_offset = j * filters * spatial;
        for (int k = 0; k < spatial; ++k) {
            sum_delta += delta[batch_offset + i * spatial + k]; // Accumulate sum
        }
    }

    mean_delta[i] = sum_delta * (-1.0f / sqrtf(variance[i] + 0.00001f)); // Compute mean_delta
}