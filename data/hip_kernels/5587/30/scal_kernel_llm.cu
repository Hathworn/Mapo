#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Simplify 1D grid index
    int stride = gridDim.x * blockDim.x;              // Calculate stride for looping

    for (int i = idx; i < N; i += stride)             // Loop over elements in increments of stride
    {
        X[i * INCX] *= ALPHA;                         // Scale the element by ALPHA
    }
}