#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateExCovX(double *e_x_cov_x, double gauss_d2, int valid_voxel_num)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Unroll loop to reduce loop overhead
    for (int i = id; i < valid_voxel_num; i += stride * 4) {
        e_x_cov_x[i] *= gauss_d2;
        if (i + stride < valid_voxel_num) e_x_cov_x[i + stride] *= gauss_d2;
        if (i + 2 * stride < valid_voxel_num) e_x_cov_x[i + 2 * stride] *= gauss_d2;
        if (i + 3 * stride < valid_voxel_num) e_x_cov_x[i + 3 * stride] *= gauss_d2;
    }
}