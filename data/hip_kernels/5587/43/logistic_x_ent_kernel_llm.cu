#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Use blockDim.x to iterate over chunks of data per block, handling more elements
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = gridDim.x * blockDim.x;
    
    for(int i = index; i < n; i += gridStride) {
        float t = truth[i];
        float p = pred[i];
        error[i] = -t * log(p + 1e-7f) - (1.0f - t) * log(1.0f - p + 1e-7f); // Use more readable exponential notation
        delta[i] = t - p; // Reduce data dependency in operations
    }
}