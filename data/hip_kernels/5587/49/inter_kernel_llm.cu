#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockDim.x * gridDim.x * blockIdx.y;
    
    // Check boundary condition only once for efficiency
    if(i >= (NX + NY) * B) return;

    int b = i / (NX + NY);
    int j = i % (NX + NY);

    // Use a conditional operator for concise code
    OUT[i] = (j < NX) ? X[b * NX + j] : Y[b * NY + j - NX];
}