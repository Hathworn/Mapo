#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify thread index calculation

    if(i < n){
        float dc_i = dc[i]; // Cache dc[i] value
        float s_i = s[i];   // Cache s[i] value

        if (da) da[i] += dc_i * s_i;
        if (db) db[i] += dc_i * (1.0f - s_i); // Use 1.0f for float operation
        ds[i] += dc_i * (a[i] - b[i]); // Combine operations for ds
    }
}