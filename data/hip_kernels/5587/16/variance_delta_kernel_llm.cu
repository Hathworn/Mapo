#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate index within grid
    if (i >= filters) return;
    
    float sum = 0.0f;  // Accumulator for variance_delta[i]
    for(int j = 0; j < batch; ++j){
        for(int k = 0; k < spatial; ++k){
            int index = j * filters * spatial + i * spatial + k;
            sum += delta[index] * (x[index] - mean[i]);  // Accumulate results
        }
    }
    
    // Store final result after complete accumulation
    variance_delta[i] = sum * -.5f * rsqrtf(variance[i] + .00001f) * rsqrtf(variance[i] + .00001f);  // Optimized using reciprocal sqrt
}