#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta) {
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;
    int b, i;
    
    // Utilize shared memory for intermediate sums
    extern __shared__ float shared_mem[];
    float *shared_output = shared_mem;

    for (b = 0; b < batch; ++b) {
        float batch_norm1 = 0;
        float batch_norm2 = 0;
        float batch_sum = 0;

        for (i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            shared_output[i] = output[i1];
            shared_output[size + i] = output[i2];

            batch_norm1 += shared_output[i] * shared_output[i];
            batch_norm2 += shared_output[size + i] * shared_output[size + i];
            batch_sum += shared_output[i] * shared_output[size + i];
        }

        norm1 += batch_norm1;
        norm2 += batch_norm2;
        sum += batch_sum;
    }

    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    sum = sum / norm;

    for (b = 0; b < batch; ++b) {
        for (i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            delta[i1] += -scale * sum * output[i2] / norm;
            delta[i2] += -scale * sum * output[i1] / norm;
        }
    }
}