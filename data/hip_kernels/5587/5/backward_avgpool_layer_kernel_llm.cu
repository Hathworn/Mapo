#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int k = id % c;
    int b = id / c;
    int out_index = k + c * b;
    float out_value = out_delta[out_index] / (w * h); // Precompute division

    for (int i = 0; i < w * h; ++i) {
        int in_index = i + w * h * out_index; // Optimize index calculation
        atomicAdd(&in_delta[in_index], out_value); // Use atomicAdd for thread safety
    }
}