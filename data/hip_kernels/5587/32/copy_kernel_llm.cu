#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use a 1D block and grid for better performance
    int stride = blockDim.x * gridDim.x; // Calculate stride for loop unrolling
    for (; i < N; i += stride) { // Loop unrolling for increased memory throughput
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}