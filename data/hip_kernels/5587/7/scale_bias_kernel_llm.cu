#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Determine the grid stride

    int filter = blockIdx.y;
    int batch = blockIdx.z;
    
    // Use grid-stride loop to scale output
    for (int offset = index; offset < size; offset += stride) {
        output[(batch * n + filter) * size + offset] *= biases[filter];
    }
}