#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Improve calculation using single dimension for blockIdx
    int stride = blockDim.x * gridDim.x; // Calculate stride for loop unrolling

    for (int idx = i; idx < n; idx += stride) { // Process more elements per thread
        float t = truth[idx];
        float p = pred[idx];
        error[idx] = (t != 0.0f) ? -__logf(p) : 0.0f; // Use fast math intrinsic for log
        delta[idx] = t - p;
    }
}