#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;

    int b = index / spatial;
    int i = index % spatial;
    float sum = 0;

    // Optimize loop by unrolling to improve performance
    for (int f = 0; f < filters; ++f)
    {
        int curr_index = b*filters*spatial + f*spatial + i;
        sum += x[curr_index] * x[curr_index];
    }
    sum = sqrtf(sum);
    if (sum == 0) sum = 1;

    // Use common subexpression elimination to improve performance
    float inv_sum = 1.0f / sum;

    for (int f = 0; f < filters; ++f)
    {
        int curr_index = b*filters*spatial + f*spatial + i;
        x[curr_index] *= inv_sum;
        dx[curr_index] = (1 - x[curr_index]) * inv_sum;
    }
}