#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for grid-stride loop

    // Use grid-stride loop for better load balancing
    for (int i = index; i < N; i += stride)
    {
        int f = (i/spatial)%filters;
        x[i] = (x[i] - mean[f])/(sqrtf(variance[f] + .00001f));
    }
}