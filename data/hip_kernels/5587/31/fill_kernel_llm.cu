#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate unique thread index in grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate total stride for grid-stride loop

    for (int i = idx; i < N; i += stride) { // Loop over all elements with step = stride
        X[i * INCX] = ALPHA; // Assign ALPHA to each element
    }
}