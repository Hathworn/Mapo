#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Shared memory for parallel reduction
    __shared__ float shared_mean;

    // Calculate mean using parallel reduction
    float sum = 0.0f;
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        sum += fabsf(weights[f * size + i]);
    }

    atomicAdd(&shared_mean, sum);  // Accumulate sum
    __syncthreads();  // Ensure all threads have updated shared_mean

    if (threadIdx.x == 0) {
        shared_mean /= size;
    }
    __syncthreads();  // Ensure mean is available for all threads

    // Binarize weights
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? shared_mean : -shared_mean;
    }
}
```
