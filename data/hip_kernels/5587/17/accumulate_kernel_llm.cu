#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Calculate the global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Initialize sum to zero
    float temp_sum = 0.0f;

    // Use shared memory to reduce multiple accesses to global memory
    extern __shared__ float shared_x[];

    // Each thread loads its portion of data from global memory to shared memory
    for (int k = threadIdx.x; k < n * groups; k += blockDim.x) {
        shared_x[threadIdx.x] = x[k];
    }
    __syncthreads();

    // Accumulate the sum
    for (int k = 0; k < n; ++k) {
        temp_sum += shared_x[k * groups + i];
    }

    // Write the result back to global memory
    sum[i] = temp_sum;
}