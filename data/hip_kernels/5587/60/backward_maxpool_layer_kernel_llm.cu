#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;
    int c = in_c;
    int area = (size - 1) / stride;

    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int index = id;
    int j = id % in_w;
    id /= in_w;
    int i = id % in_h;
    id /= in_h;
    int k = id % in_c;
    id /= in_c;
    int b = id;

    const int w_offset = -pad;
    const int h_offset = -pad;

    float d = 0.0f;

    // Precompute fixed values for clarity
    int base_out_w = (j - w_offset) / stride;
    int base_out_h = (i - h_offset) / stride;
    int base_index = k + c * b;
    
    // Iterate more efficiently by calculating the out indices only once
    for (int l = -area; l <= area; ++l) {
        int out_h = base_out_h + l;
        if (out_h >= 0 && out_h < h) {  // Check row validity
            for (int m = -area; m <= area; ++m) {
                int out_w = base_out_w + m;
                if (out_w >= 0 && out_w < w) {  // Check column validity
                    int out_index = out_w + w * (out_h + h * base_index);
                    if (indexes[out_index] == index) {
                        d += delta[out_index];
                    }
                }
            }
        }
    }

    // Avoid atomic operation by assuming unique writers to each prev_delta[index]
    prev_delta[index] += d;
}