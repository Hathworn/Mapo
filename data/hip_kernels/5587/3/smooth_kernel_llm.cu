#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int w_offset = -(size / 2.f);
    int h_offset = -(size / 2.f);

    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    int out_index = j + w * (i + h * (k + c * b));
    float delta_accum = 0;  // Accumulates delta for atomic update

    for (int l = 0; l < size; ++l) {
        int cur_h = h_offset + i + l;
        if (cur_h < 0 || cur_h >= h) continue;  // Early exit if out of bounds

        for (int m = 0; m < size; ++m) {
            int cur_w = w_offset + j + m;
            if (cur_w < 0 || cur_w >= w) continue;  // Early exit if out of bounds

            int index = cur_w + w * (cur_h + h * (k + b * c));
            delta_accum += rate * (x[index] - x[out_index]);
        }
    }

    // Atomic add to handle concurrent writes
    atomicAdd(&delta[out_index], delta_accum);
}