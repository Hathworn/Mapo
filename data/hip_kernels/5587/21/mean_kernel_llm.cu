#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Calculate a reciprocal to avoid repeated division.
    const float scale = 1.f / (batch * spatial);
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Return early if out of bounds
    if (i >= filters) return;
    
    float sum = 0.0f; // Use a local variable for accumulation

    // Combine loops to improve data locality and reduce index calculations
    for (int j = 0; j < batch * spatial; ++j) {
        int index = j * filters + i;
        sum += x[index]; // Accumulate directly into local variable
    }
    
    mean[i] = sum * scale; // Assign once after multiplication
}