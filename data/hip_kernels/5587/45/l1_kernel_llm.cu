#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if(idx < n) {
        float diff = truth[idx] - pred[idx];
        error[idx] = fabsf(diff); // Use faster fabsf for float absolute value
        delta[idx] = copysignf(1.0f, diff); // Use copysignf to determine the sign
    }
}