#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use one-dimensional grid and streamline index calculations
    if (i < n) {
        // Optimize computation by directly accessing global index
        c[i] += a[i] * b[i];
    }
}