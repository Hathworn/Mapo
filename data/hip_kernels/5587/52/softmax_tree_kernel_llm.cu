#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output) 
{
    extern __shared__ float shared_output[]; // Shared memory buffer for intermediate output values
    int i;
    float sum = 0;
    float largest = -INFINITY;
    // Optimize loop with stride increment and shared memory
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        float val = input[i * stride];
        atomicMax_block(&largest, val); // Using atomic operation to find the maximum
    }
    __syncthreads(); // Ensure all threads have computed their largest
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        float e = expf(input[i * stride] / temp - largest / temp);
        atomicAdd_block(&sum, e); // Atomic add to accumulate the sum
        shared_output[i * stride] = e; // Store intermediate output
    }
    __syncthreads(); // Wait for all threads to compute their exp
    for (i = threadIdx.x; i < n; i += blockDim.x) {
        output[i * stride] = shared_output[i * stride] / sum; // Normalizing the output
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset) 
{
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;
    int s = id % spatial;
    id = id / spatial;
    int g = id % groups;
    int b = id / groups;
    int goff = group_offset[g] * spatial;
    int boff = b * stride;
    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}
```
