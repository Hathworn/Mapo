#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Using shared memory to cache mean values
    extern __shared__ float shared_mean[];
    int tid = threadIdx.x;

    // Calculate global thread index
    int global_idx = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + tid;
    if (tid < filters) {
        shared_mean[tid] = mean[tid];
    }
    __syncthreads();

    if (global_idx >= filters) return;

    float var = 0.0f;
    float scale = 1.f / (batch * spatial - 1);

    // Unrolling the loops to optimize performance
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; k += 4) {
            int index = j * filters * spatial + global_idx * spatial + k;
            float tmp1 = (index < batch * filters * spatial) ? x[index] - shared_mean[global_idx] : 0;
            float tmp2 = (index + 1 < batch * filters * spatial) ? x[index + 1] - shared_mean[global_idx] : 0;
            float tmp3 = (index + 2 < batch * filters * spatial) ? x[index + 2] - shared_mean[global_idx] : 0;
            float tmp4 = (index + 3 < batch * filters * spatial) ? x[index + 3] - shared_mean[global_idx] : 0;
            var += tmp1 * tmp1 + tmp2 * tmp2 + tmp3 * tmp3 + tmp4 * tmp4;
        }
    }
    variance[global_idx] = var * scale;
}