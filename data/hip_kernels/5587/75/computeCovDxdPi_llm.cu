#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computeCovDxdPi(int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *inverse_covariance, int voxel_num, double gauss_d1, double gauss_d2, double *point_gradients, double *cov_dxd_pi, int valid_voxel_num)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int row = blockIdx.y;
    int col = blockIdx.z;

    // Process only necessary rows and columns
    if (row < 3 && col < 6) {
        double *icov0 = inverse_covariance + row * 3 * voxel_num;
        double *icov1 = icov0 + voxel_num;
        double *icov2 = icov1 + voxel_num;
        double *cov_dxd_pi_tmp = cov_dxd_pi + (row * 6 + col) * valid_voxel_num;
        double *pg_tmp0 = point_gradients + col * valid_points_num;
        double *pg_tmp1 = pg_tmp0 + 6 * valid_points_num;
        double *pg_tmp2 = pg_tmp1 + 6 * valid_points_num;

        // Unroll loop for better performance
        for (int i = id; i < valid_points_num; i += stride) {
            double pg0 = pg_tmp0[i];
            double pg1 = pg_tmp1[i];
            double pg2 = pg_tmp2[i];

            // Efficient memory access by storing index once
            int start_voxel = starting_voxel_id[i];
            const int end_voxel = starting_voxel_id[i + 1];

            // Use one-load-per-access pattern for voxel ids
            for (int j = start_voxel; j < end_voxel; j++) {
                int vid = voxel_id[j];

                cov_dxd_pi_tmp[j] = icov0[vid] * pg0 + icov1[vid] * pg1 + icov2[vid] * pg2;
            }
        }
    }
}