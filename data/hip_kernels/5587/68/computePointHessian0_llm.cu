#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computePointHessian0(float *x, float *y, float *z, int points_num, int *valid_points, int valid_points_num, double *dh_ang, double *ph93, double *ph103, double *ph113, double *ph123, double *ph94, double *ph133, double *ph104, double *ph143, double *ph114, double *ph153, double *ph95, double *ph163, double *ph105, double *ph173, double *ph115)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    __shared__ double h_ang[18];

    // Load dh_ang into shared memory for faster access
    if (threadIdx.x < 18) {
        h_ang[threadIdx.x] = dh_ang[threadIdx.x];
    }

    __syncthreads();

    for (int i = id; i < valid_points_num; i += stride) {
        int pid = valid_points[i];

        // Use registers for computations to reduce memory access
        double o_x = static_cast<double>(x[pid]);
        double o_y = static_cast<double>(y[pid]);
        double o_z = static_cast<double>(z[pid]);

        // Pre-compute values used multiple times to avoid redundant calculations
        double t1 = o_x * h_ang[0] + o_y * h_ang[1] + o_z * h_ang[2];
        double t2 = o_x * h_ang[3] + o_y * h_ang[4] + o_z * h_ang[5];
        double t3 = o_x * h_ang[6] + o_y * h_ang[7] + o_z * h_ang[8];
        double t4 = o_x * h_ang[9] + o_y * h_ang[10] + o_z * h_ang[11];
        double t5 = o_x * h_ang[12] + o_y * h_ang[13] + o_z * h_ang[14];
        double t6 = o_x * h_ang[15] + o_y * h_ang[16] + o_z * h_ang[17];

        ph93[i] = 0;
        ph103[i] = t1;
        ph113[i] = t2;

        ph123[i] = ph94[i] = 0;
        ph133[i] = ph104[i] = t3;
        ph143[i] = ph114[i] = t4;

        ph153[i] = ph95[i] = 0;
        ph163[i] = ph105[i] = t5;
        ph173[i] = ph115[i] = t6;
    }
}