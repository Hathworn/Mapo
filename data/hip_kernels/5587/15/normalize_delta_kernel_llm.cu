#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    // Calculate the global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for variance_delta and mean to reduce global memory accesses
    __shared__ float shared_variance_delta;
    __shared__ float shared_mean;
    
    if (index < N) {
        int f = (index/spatial) % filters;

        // load variance_delta and mean into shared memory for faster access
        if (threadIdx.x == 0) {
            shared_variance_delta = variance_delta[f];
            shared_mean = mean[f];
        }
        __syncthreads();

        // Perform computation with cached shared values
        float norm_variance = 1.f / (sqrtf(variance[f] + 0.00001f));
        delta[index] = delta[index] * norm_variance +
                       shared_variance_delta * 2.f * (x[index] - shared_mean) / (spatial * batch) +
                       mean_delta[f] / (spatial * batch);
    }
}