#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x;
    float s = hsv.y;
    float v = hsv.z;

    float r, g, b;
    float f, p, q, t;

    if (s == 0) {
        r = g = b = v;
    } else {
        int index = (int) floorf(h * 6.0f); // Normalize h to 0-6 range
        f = h * 6.0f - index; // Updated fractional part of h
        p = v * (1.0f - s);
        q = v * (1.0f - s * f);
        t = v * (1.0f - s * (1.0f - f));

        switch (index % 6) { // Use modulus for cyclic indexing
            case 0: r = v; g = t; b = p; break;
            case 1: r = q; g = v; b = p; break;
            case 2: r = p; g = v; b = t; break;
            case 3: r = p; g = q; b = v; break;
            case 4: r = t; g = p; b = v; break;
            default: r = v; g = p; b = q; break;
        }
    }
    return make_float3(fmaxf(0.0f, fminf(1.0f, r)), 
                       fmaxf(0.0f, fminf(1.0f, g)), 
                       fmaxf(0.0f, fminf(1.0f, b)));
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x;
    float g = rgb.y;
    float b = rgb.z;

    float h, s, v;
    float max = fmaxf(fmaxf(r, g), b);
    float min = fminf(fminf(r, g), b);
    float delta = max - min;
    v = max;
    
    if(max == 0){
        s = 0;
        h = -1;
    } else {
        s = delta / max;
        if(r == max) h = (g - b) / delta;
        else if(g == max) h = 2.0f + (b - r) / delta;
        else h = 4.0f + (r - g) / delta;
        h /= 6.0f; // Normalize h to 0-1 range
        if (h < 0) h += 1.0f;
    }

    return make_float3(h, s, v);
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= size) return;

    // Precompute offsets and index conversions
    int x = id % w;
    int y = id / w % h;
    int batchIndex = id / (w * h);

    float rshift = rand[0];
    float gshift = rand[1];
    float bshift = rand[2];
    float r0 = rand[8 * batchIndex];
    float r1 = rand[8 * batchIndex + 1];
    float r2 = rand[8 * batchIndex + 2];
    float r3 = rand[8 * batchIndex + 3];

    saturation = r0 * (saturation - 1.0f) + 1.0f;
    saturation = (r1 > 0.5f) ? 1.0f / saturation : saturation;
    exposure = r2 * (exposure - 1.0f) + 1.0f;
    exposure = (r3 > 0.5f) ? 1.0f / exposure : exposure;

    size_t offset = batchIndex * h * w * 3;
    image += offset;

    float r = image[x + w * (y + h * 0)];
    float g = image[x + w * (y + h * 1)];
    float b = image[x + w * (y + h * 2)];
    float3 rgb = make_float3(r, g, b);

    if(train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }

    // Scale and shift RGB values
    image[x + w * (y + h * 0)] = rgb.x * scale + translate + (rshift - 0.5f) * shift;
    image[x + w * (y + h * 1)] = rgb.y * scale + translate + (gshift - 0.5f) * shift;
    image[x + w * (y + h * 2)] = rgb.z * scale + translate + (bshift - 0.5f) * shift;
}