#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    // Calculate global thread index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Ensure all threads coalesce on conditional checks
    if (i < n) {
        float mask_val = mask[i]; // Reduce indexing operations
        if (mask_val == mask_num) {
            x[i] = val;
        }
    }
}