#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= N) return;
    
    // Precompute repeated calculations
    int in_c_step = w * h;
    int batch_step = in_c_step * c;
    int stride2 = stride * stride;
    
    int in_index = i;
    int in_w = i % w;
    i /= w;
    int in_h = i % h;
    i /= h;
    int in_c = i % c;
    int b = i / c;
    
    int out_c = c / stride2;
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + (offset % stride);
    int h2 = in_h * stride + (offset / stride);
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));
    
    if (forward)
        out[out_index] = x[in_index];
    else
        out[in_index] = x[out_index];
}