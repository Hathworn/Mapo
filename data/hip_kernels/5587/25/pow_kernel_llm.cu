#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Calculate linear thread index within the grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Remove unnecessary computation, blockIdx.y*gridDim.x, to optimize performance
    if (idx < N)
    {
        Y[idx * INCY] = powf(X[idx * INCX], ALPHA); // Use powf() for single precision
    }
}