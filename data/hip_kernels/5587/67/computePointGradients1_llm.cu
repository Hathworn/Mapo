#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize by using loop unrolling and reducing shared memory accesses
__global__ void computePointGradients1(float *x, float *y, float *z, int points_num, int *valid_points, int valid_points_num, double *dj_ang, double *pg24, double *pg05, double *pg15, double *pg25)
{
    __shared__ double j_ang[12];
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    if (threadIdx.x < 12) {
        j_ang[threadIdx.x] = dj_ang[threadIdx.x + 12];
    }

    __syncthreads();

    for (int i = id; i < valid_points_num; i += stride) {
        int pid = valid_points[i];

        // Orignal coordinates
        double o_x = static_cast<double>(x[pid]);
        double o_y = static_cast<double>(y[pid]);
        double o_z = static_cast<double>(z[pid]);

        // Compute point derivatives using loop unrolling for efficiency
        double j_ang0 = j_ang[0], j_ang1 = j_ang[1], j_ang2 = j_ang[2];
        pg24[i] = o_x * j_ang0 + o_y * j_ang1 + o_z * j_ang2;

        double j_ang3 = j_ang[3], j_ang4 = j_ang[4], j_ang5 = j_ang[5];
        pg05[i] = o_x * j_ang3 + o_y * j_ang4 + o_z * j_ang5;

        double j_ang6 = j_ang[6], j_ang7 = j_ang[7], j_ang8 = j_ang[8];
        pg15[i] = o_x * j_ang6 + o_y * j_ang7 + o_z * j_ang8;

        double j_ang9 = j_ang[9], j_ang10 = j_ang[10], j_ang11 = j_ang[11];
        pg25[i] = o_x * j_ang9 + o_y * j_ang10 + o_z * j_ang11;
    }
}