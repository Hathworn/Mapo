#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void computePointHessian1(float *x, float *y, float *z, int points_num, int *valid_points, int valid_points_num, double *dh_ang, double *ph124, double *ph134, double *ph144, double *ph154, double *ph125, double *ph164, double *ph135, double *ph174, double *ph145)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    __shared__ double h_ang[18];

    if (threadIdx.x < 18) {
        h_ang[threadIdx.x] = dh_ang[18 + threadIdx.x];
    }

    __syncthreads();

    for (int i = id; i < valid_points_num; i += stride) {
        int pid = valid_points[i];

        // Original coordinates
        double o_x = static_cast<double>(x[pid]);
        double o_y = static_cast<double>(y[pid]);
        double o_z = static_cast<double>(z[pid]);

        double hx = o_x * h_ang[0] + o_y * h_ang[1] + o_z * h_ang[2];
        double hy = o_x * h_ang[3] + o_y * h_ang[4] + o_z * h_ang[5];
        double hz = o_x * h_ang[6] + o_y * h_ang[7] + o_z * h_ang[8];

        // Write results
        ph124[i] = hx;
        ph134[i] = hy;
        ph144[i] = hz;

        hx = o_x * h_ang[9] + o_y * h_ang[10] + o_z * h_ang[11];
        hy = o_x * h_ang[12] + o_y * h_ang[13] + o_z * h_ang[14];
        hz = o_x * h_ang[15] + o_y * h_ang[16] + o_z * h_ang[17];

        ph154[i] = ph125[i] = hx;
        ph164[i] = ph135[i] = hy;
        ph174[i] = ph145[i] = hz;
    }
}