#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index based on the optimized 2D grid and block configuration.
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we process only valid elements within the matrix.
    if (i < N) 
    {
        // Perform the addition operation directly.
        X[i * INCX] += ALPHA;
    }
}