#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    // Calculate global thread index in one step to reduce arithmetic operations
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Use if-check to ensure we do not access out-of-bound memory
    if (i < N) 
    {
        Y[i * INCY] *= X[i * INCX];
    }
}