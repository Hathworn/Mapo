#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate the global index of the thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for better GPU utilization
    for(int idx = i; idx < n; idx += blockDim.x * gridDim.x) {
        // Perform the weighted sum computation
        c[idx] = s[idx] * a[idx] + (1 - s[idx]) * (b ? b[idx] : 0);
    }
}