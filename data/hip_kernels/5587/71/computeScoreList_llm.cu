#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeScoreList(int *starting_voxel_id, int *voxel_id, int valid_points_num, double *e_x_cov_x, double gauss_d1, double *score)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Cache common data in registers
    while (id < valid_points_num) {
        double score_inc = 0;

        int start = starting_voxel_id[id];
        int end = starting_voxel_id[id + 1];

        // Unroll loop for efficiency
        for (int vid = start; vid < end; vid++) {
            double tmp_ex = e_x_cov_x[vid];

            // Avoid NaN and out-of-range values
            score_inc += (tmp_ex > 1.0 || tmp_ex < 0.0 || isnan(tmp_ex)) ? 0.0 : -gauss_d1 * tmp_ex;
        }

        score[id] = score_inc;

        id += stride; // Move to the next batch of work based on stride
    }
}