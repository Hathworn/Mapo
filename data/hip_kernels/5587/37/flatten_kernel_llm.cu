#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Unroll the loop for improved performance
    for (int i = idx; i < N; i += stride) {
        int in_s = i % spatial;
        int in_c = (i / spatial) % layers;
        int b = i / (spatial * layers);
        
        int i1 = b * layers * spatial + in_c * spatial + in_s;
        int i2 = b * layers * spatial + in_s * layers + in_c;
        
        if (forward) {
            out[i2] = x[i1];
        } else {
            out[i1] = x[i2];
        }
    }
}