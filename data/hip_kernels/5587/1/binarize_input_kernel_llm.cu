#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = blockIdx.x * blockDim.x + threadIdx.x; // Optimize: Remove unnecessary blockIdx.y and gridDim.x calculations
    if (s >= size) return;

    float mean = 0.0f;
    for (int i = 0; i < n; ++i){
        mean += fabsf(input[i * size + s]);
    }
    mean /= n;

    float threshold = mean; // Optimize: Cache the computed mean value for reuse
    for (int i = 0; i < n; ++i){
        float inputValue = input[i * size + s];
        binary[i * size + s] = (inputValue > 0.0f) ? threshold : -threshold;
    }
}