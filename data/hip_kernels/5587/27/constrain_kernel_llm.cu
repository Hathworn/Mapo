#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride of total threads across the grid for memory access pattern improvement
    int stride = blockDim.x * gridDim.x; 
    
    // Loop over the array with appropriate stride
    for(; i < N; i += stride) {
        X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i * INCX]));
    }
}