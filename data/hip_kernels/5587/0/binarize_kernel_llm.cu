#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Optimize thread index calculation and reduce global memory accesses
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        binary[i] = (x[i] >= 0.0f) ? 1.0f : -1.0f;
    }
}