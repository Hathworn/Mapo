#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    // Calculate flattened unique index for each thread
    int index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (index >= N) return;

    // Using pre-computed and shared memory for efficiency
    float one_minus_B1_t = 1.f - powf(B1, t);
    float one_minus_B2_t = 1.f - powf(B2, t);
    
    float mhat = m[index] / one_minus_B1_t;
    float vhat = v[index] / one_minus_B2_t;
    
    // Efficient update of parameters
    x[index] += rate * mhat / (sqrtf(vhat) + eps);
}