#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Remove unnecessary gridIdx.y calculation

    if (i < N) {
        float val = X[i * INCX];  // Load value once and use a register for it
        if (val * val < ALPHA * ALPHA) {
            X[i * INCX] = 0;
        }
    }
}