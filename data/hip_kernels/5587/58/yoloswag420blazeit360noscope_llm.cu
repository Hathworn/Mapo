#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    // Calculate global thread ID with optimizations
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if within bounds and perform operation
    if(id < size) {
        float rnd = rand[id];
        float inp = input[id];
        input[id] = (rnd < prob) ? 0.0f : inp * scale;  // Use temporary variables for better readability
    }
}