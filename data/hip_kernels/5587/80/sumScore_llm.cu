#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumScore(double *score, int full_size, int half_size)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Ensure threads do not go out of bounds
    if (index < half_size) {
        // Apply stride only if necessary
        for (int i = index; i < half_size; i += stride) {
            // Use shared memory for partial scores 
            score[i] += (i + half_size < full_size) ? score[i + half_size] : 0;
        }
    }
}