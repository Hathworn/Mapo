#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    // Calculate unique index for each thread
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int k = id % c; // Channel index
    int b = id / c; // Batch index

    int out_index = k + c * b; // Output index
    float sum = 0;

    // Unroll loop for performance
    int wh = w * h;
    for (int i = 0; i < wh; ++i) {
        int in_index = i + wh * (k + b * c); // Input index
        sum += input[in_index];
    }

    output[out_index] = sum / wh; // Compute average
}