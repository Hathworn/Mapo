#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);

        // Use ternary operator to simplify conditional assignment
        error[i] = (abs_val < 1) ? diff * diff : 2 * abs_val - 1;
        delta[i] = (abs_val < 1) ? diff : (diff > 0) ? 1 : -1;
    }
}