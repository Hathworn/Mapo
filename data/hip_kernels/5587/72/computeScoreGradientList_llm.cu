#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeScoreGradientList(float *trans_x, float *trans_y, float *trans_z, int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *centroid_x, double *centroid_y, double *centroid_z, int voxel_num, double *e_x_cov_x, double *cov_dxd_pi, double gauss_d1, int valid_voxel_num, double *score_gradients)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;
    int col = blockIdx.y;

    if (col < 6) {
        double *sg = score_gradients + col * valid_points_num;
        double *cov_dxd_pi_mat0 = cov_dxd_pi + col * valid_voxel_num;
        double *cov_dxd_pi_mat1 = cov_dxd_pi_mat0 + 6 * valid_voxel_num;
        double *cov_dxd_pi_mat2 = cov_dxd_pi_mat1 + 6 * valid_voxel_num;

        for (int i = id; i < valid_points_num; i += stride) {
            int pid = valid_points[i];
            double d_x = static_cast<double>(trans_x[pid]);
            double d_y = static_cast<double>(trans_y[pid]);
            double d_z = static_cast<double>(trans_z[pid]);

            double tmp_sg = 0.0;

            // Cache current centroid coordinates for better memory access pattern
            int start_vid = starting_voxel_id[i];
            int end_vid = starting_voxel_id[i + 1];
            
            for (int j = start_vid; j < end_vid; j++) {
                int vid = voxel_id[j];
                double tmp_ex = e_x_cov_x[j];

                if (!(tmp_ex > 1 || tmp_ex < 0 || tmp_ex != tmp_ex)) {
                    tmp_ex *= gauss_d1;

                    // Use shared memory or register variables where possible to reduce global memory access
                    double dx = d_x - centroid_x[vid];
                    double dy = d_y - centroid_y[vid];
                    double dz = d_z - centroid_z[vid];

                    tmp_sg += (dx * cov_dxd_pi_mat0[j] + dy * cov_dxd_pi_mat1[j] + dz * cov_dxd_pi_mat2[j]) * tmp_ex;
                }
            }

            sg[i] = tmp_sg;
        }
    }
}