#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + 2*pad) / stride;
    int w = (in_w + 2*pad) / stride;
    int c = in_c;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    int b = id / (w * h * c);
    int k = (id / (w * h)) % c;
    int i = (id / w) % h;
    int j = id % w;

    int out_index = j + w * (i + h * (k + c * b));
    float max = -INFINITY;
    int max_i = -1;
    int w_offset = -pad + j * stride;
    int h_offset = -pad + i * stride;

    // Loop unrolling for smaller `size` and precompute offsets
    for (int l = 0; l < size; ++l) {
        int cur_h = h_offset + l;
        if (cur_h < 0 || cur_h >= in_h) continue;
        for (int m = 0; m < size; ++m) {
            int cur_w = w_offset + m;
            if (cur_w < 0 || cur_w >= in_w) continue;
            int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
            float val = input[index];
            if (val > max) {
                max = val;
                max_i = index;
            }
        }
    }
    output[out_index] = max;
    indexes[out_index] = max_i;
}