#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Use grid-stride loop for better scalability
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        float diff = truth[i] - pred[i];
        error[i] = diff * diff; // Calculate squared difference
        delta[i] = diff;        // Store difference
    }
}