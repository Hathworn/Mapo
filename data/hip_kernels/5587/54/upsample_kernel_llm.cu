#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (i >= N) return;

    int out_index = i;
    int out_w = out_index % (w * stride);
    int out_h = (out_index / (w * stride)) % (h * stride);
    int out_c = (out_index / (w * stride) / (h * stride)) % c;
    int b = out_index / (w * stride) / (h * stride) / c;

    int in_w = out_w / stride;
    int in_h = out_h / stride;

    int in_index = b * w * h * c + out_c * w * h + in_h * w + in_w;

    if (forward) {
        out[out_index] += scale * x[in_index]; // Forward pass
    } else {
        atomicAdd(&x[in_index], scale * out[out_index]); // Backward pass with atomic operation
    }
}