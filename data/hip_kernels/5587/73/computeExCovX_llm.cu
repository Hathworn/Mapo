#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeExCovX(float *trans_x, float *trans_y, float *trans_z, int *valid_points, int *starting_voxel_id, int *voxel_id, int valid_points_num, double *centr_x, double *centr_y, double *centr_z, double gauss_d1, double gauss_d2, double *e_x_cov_x, double *icov00, double *icov01, double *icov02, double *icov10, double *icov11, double *icov12, double *icov20, double *icov21, double *icov22)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Loop unrolling to reduce loop overhead
    for (int i = id; i < valid_points_num; i += stride) {
        int pid = valid_points[i];
        double d_x = static_cast<double>(trans_x[pid]);
        double d_y = static_cast<double>(trans_y[pid]);
        double d_z = static_cast<double>(trans_z[pid]);

        for (int j = starting_voxel_id[i]; j < starting_voxel_id[i + 1]; j++) {
            int vid = voxel_id[j];

            double t_x = d_x - centr_x[vid];
            double t_y = d_y - centr_y[vid];
            double t_z = d_z - centr_z[vid];

            // Load data into registers to reduce memory access time
            double icov00_vid = icov00[vid];
            double icov01_vid = icov01[vid];
            double icov02_vid = icov02[vid];
            double icov10_vid = icov10[vid];
            double icov11_vid = icov11[vid];
            double icov12_vid = icov12[vid];
            double icov20_vid = icov20[vid];
            double icov21_vid = icov21[vid];
            double icov22_vid = icov22[vid];

            // Minimize repeated calculations and use fused multiply-add
            double exp_arg = gauss_d2 * ((t_x * icov00_vid + t_y * icov01_vid + t_z * icov02_vid) * t_x
                         + (t_x * icov10_vid + t_y * icov11_vid + t_z * icov12_vid) * t_y
                         + (t_x * icov20_vid + t_y * icov21_vid + t_z * icov22_vid) * t_z) / 2.0;

            e_x_cov_x[j] = exp(-exp_arg);
        }
    }
}