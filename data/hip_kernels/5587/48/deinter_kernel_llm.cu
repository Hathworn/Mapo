#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified calculation for 'i'
    int b = blockIdx.y; // Separate dimension for batches

    if(i < NX + NY) { // Reduced boundary check
        int idx = b * (NX + NY) + i; // Calculate full offset
        if (i < NX) {
            if (X) X[b * NX + i] += OUT[idx]; // Use calculated idx
        } else {
            if (Y) Y[b * NY + i - NX] += OUT[idx]; // Use calculated idx
        }
    }
}