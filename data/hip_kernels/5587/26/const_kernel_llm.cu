#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index based on 1D grid and block
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop to cover all elements efficiently
    for (int i = idx; i < N; i += blockDim.x * gridDim.x)
    {
        X[i * INCX] = ALPHA;
    }
}