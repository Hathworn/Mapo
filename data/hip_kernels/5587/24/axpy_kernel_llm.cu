#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Use a more efficient calculation for global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = gridDim.x * blockDim.x;
    
    // Stride through the entire vector while keeping the number of kernels low
    for (; i < N; i += gridStride) 
    {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}