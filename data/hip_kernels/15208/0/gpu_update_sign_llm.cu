#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ void set_shared(int *buff, int *G, int off1, int off2, int n) {
    int m = blockIdx.x + off1 * gridDim.x;
    int l = threadIdx.x + off2 * blockDim.x;
    int maxx = blockDim.x - 1;

    if (m < n && l < n) {
        if (l == n - 1) {
            if (blockDim.x > n)
                maxx = n - 1;
            else if (n / blockDim.x == off2)
                maxx = (n - 1) % blockDim.x;
        }

        if ((threadIdx.x == 0 || threadIdx.x == maxx) && maxx != 0) {
            int ad = (threadIdx.x == 0) ? -2 : 0;

            for (int i = m - 2; i <= m + 2; i++) {
                for (int j = l + ad; j <= l + ad + 2; j++) {
                    int h1 = i - m;
                    int h2 = j - l;
                    int b_ind_x = 2 + h1;
                    int b_ind_y = threadIdx.x + 2 + h2;
                    int g_ind_x = (n + i) % n;
                    int g_ind_y = (n + j) % n;
                    buff[b_ind_x * (blockDim.x + 4) + b_ind_y] = G[g_ind_x * n + g_ind_y];
                }
            }
        } else if (threadIdx.x == maxx && maxx == 0) {
            for (int i = m - 2; i <= m + 2; i++) {
                for (int j = l - 2; j <= l + 2; j++) {
                    int h1 = i - m;
                    int h2 = j - l;
                    int b_ind_x = 2 + h1;
                    int b_ind_y = threadIdx.x + 2 + h2;
                    int g_ind_x = (n + i) % n;
                    int g_ind_y = (n + j) % n;
                    buff[b_ind_x * (blockDim.x + 4) + b_ind_y] = G[g_ind_x * n + g_ind_y];
                }
            }
        } else {
            for (int i = m - 2; i <= m + 2; i++) {
                int h1 = i - m;
                int b_ind_x = 2 + h1;
                int b_ind_y = threadIdx.x + 2;
                int g_ind_x = (n + i) % n;
                int g_ind_y = (n + l) % n;
                buff[b_ind_x * (blockDim.x + 4) + b_ind_y] = G[g_ind_x * n + g_ind_y];
            }
        }
    }
}

__global__ void gpu_update_sign(int *G, double *w, int k, int n, int *temp, int *flag, int it_b, int it_t) {
    int buf = 0;
    __shared__ int buff[5140];

    for (int off1 = 0; off1 < it_b; off1++) {
        for (int off2 = 0; off2 < it_t; off2++) {
            // Optimize: Only calculate shared memory space when in bounds
            if (blockIdx.x + off1 * gridDim.x < n && threadIdx.x + off2 * blockDim.x < n) {
                set_shared(buff, G, off1, off2, n);
            }
            __syncthreads();

            int x = blockIdx.x + off1 * gridDim.x;
            int y = threadIdx.x + off2 * blockDim.x;
            if (x < n && y < n) {
                double sum = 0.0;
                for (int i = 0; i < k; i++) {
                    for (int j = 0; j < k; j++) {
                        sum += ((double)buff[i * (blockDim.x + 4) + (threadIdx.x + j)]) * w[i * k + j];
                    }
                }
                int result;
                if (sum > 1e-6) {
                    result = 1;
                    if (result != buff[2 * (blockDim.x + 4) + threadIdx.x + 2]) buf++;
                } else if (sum < -(1e-6)) {
                    result = -1;
                    if (result != buff[2 * (blockDim.x + 4) + threadIdx.x + 2]) buf++;
                } else {
                    result = buff[2 * (blockDim.x + 4) + threadIdx.x + 2];
                }
                temp[x * n + y] = result;
            }
            __syncthreads();
        }
        __syncthreads();
    }
    atomicAdd(flag, buf); // Optimize: Use atomic operation for thread-safe flag increment
}