#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_get_neighbors(int *neighbors, int n, int k) {
    // Calculate the starting index for each block and thread
    int m = blockIdx.x * (n / gridDim.x) + threadIdx.x;
    int l = blockIdx.y * (n / gridDim.y) + threadIdx.y;

    // Ensure indices are within bounds
    if (m < n && l < n) {
        int counter_i = 0;
        for (int i = m - (k / 2); i <= m + (k / 2); i++) {
            int counter_j = 0;
            for (int j = l - (k / 2); j <= l + (k / 2); j++) {
                int index = m * n * k * k + l * k * k + counter_i * k + counter_j;
                int index_i = (n + i) % n;
                int index_j = (n + j) % n;
                neighbors[index] = index_i * n + index_j;
                counter_j++;
            }
            counter_i++;
        }
    }
}