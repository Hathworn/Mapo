#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_update_sign(int *G, double *w, int *neighbors, int k, int n, int *temp, int *flag, int it_b, int it_t) {
    int result;
    double sum = 0.0;
    int buf = 0;

    // Optimize index calculations and bounds checking
    int x = blockIdx.x + it_b * gridDim.x;
    int y = threadIdx.x + it_t * blockDim.x;
    if (x < n && y < n) {

        // Move repetitive index calculation out of loops
        int baseIndex = neighbors[x * n * k * k + y * k * k];

        // Use local variable for current element
        int currentElem = G[baseIndex + 12];

        // Use registers for loop boundaries
        int kSquared = k * k;

        // Calculate result using sequential memory access
        for (int i = 0; i < k; i++) {
            for (int j = 0; j < k; j++) {
                sum += ((double)G[baseIndex + i * k + j]) * w[i * k + j];
            }
        }

        // Evaluate and write back
        if (sum > 1e-6) {
            result = 1;
        } else if (sum < -(1e-6)) {
            result = -1;
        } else {
            result = currentElem;
        }

        // Only increment flag if necessary
        if (result != currentElem) {
            buf++;
        }

        atomicAdd(flag, buf);
        temp[x * n + y] = result;
    }
}