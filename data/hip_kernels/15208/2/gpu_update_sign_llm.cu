#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_update_sign(int *G, double *w, int *neighbors, int k, int n, int *temp, int *flag, int it_b, int it_t)
{
    // Use shared memory to reduce global memory accesses
    extern __shared__ double shared_w[];
    
    // Load weights into shared memory
    int tid = threadIdx.x;
    for (int i = tid; i < k*k; i += blockDim.x) {
        shared_w[i] = w[i];
    }
    __syncthreads();
    
    int buf = 0;
    
    // Reduce loop iterations using stride technique
    int stride_b = gridDim.x * it_b;
    int stride_t = blockDim.x * it_t;
    
    for (int off1 = blockIdx.x; off1 < n; off1 += stride_b) {
        for (int off2 = threadIdx.x; off2 < n; off2 += stride_t) {
            int result;
            double sum = 0.0;
    
            int x = off1;
            int y = off2;
    
            if (x < n && y < n) {
                for (int i = 0; i < k; i++) {
                    for (int j = 0; j < k; j++) {
                        sum += ((double)G[neighbors[x*n*k*k+y*k*k+i*k+j]]) * shared_w[i*k+j];
                    }
                }
    
                if (sum > 1e-6) {
                    result = 1;
                    if (result != G[neighbors[x*n*k*k+y*k*k+12]])
                        buf++;
                } else if (sum < -(1e-6)) {
                    result = -1;
                    if (result != G[neighbors[x*n*k*k+y*k*k+12]])
                        buf++;
                } else {
                    result = G[neighbors[x*n*k*k+y*k*k+12]];
                }
                temp[x*n+y] = result;
            }
        }
    }
    
    // Use atomic addition to update the flag
    atomicAdd(flag, buf);
}