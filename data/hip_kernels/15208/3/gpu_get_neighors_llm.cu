#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_get_neighbors(int *neighbors, int n, int k) {
    // Calculate global thread index for linear access
    int m = blockIdx.x * blockDim.x + threadIdx.x;
    int l = blockIdx.y * blockDim.y + threadIdx.y;

    int counter_i = 0;
    if(m < n && l < n) {
        // Iterate over neighborhood of size k
        for (int i = m - (k / 2); i <= m + (k / 2); i++) {
            int counter_j = 0;
            for (int j = l - (k / 2); j <= l + (k / 2); j++) {
                int index = m * n * k * k + l * k * k + counter_i * k + counter_j;
                int index_i = (n + i) % n;
                int index_j = (n + j) % n;
                neighbors[index] = index_i * n + index_j;
                counter_j++;
            }
            counter_i++;
        }
    }
}