#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure thread work is within bounds
    if (tid < N) {
        // Use ternary operation for concise writing
        ingrad[tid] = (indata[tid] > 0) ? outgrad[tid] : 0.0f;
    }
}