#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop through all elements to utilize each thread effectively
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        out[i] = in[i] > 0 ? in[i] : 0;
    }
}