#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void zero_dm_outliers_kernel_two(unsigned short *d_input, int nchans, int nsamp)
{
    // Use registers instead of local variables where possible
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    float stdev = 1000000.0f, mean = MEAN, mean_last = 0.0f, sum, sum_squares, cutoff = CUT * stdev;
    int count, iters = 0;

    __shared__ float g_mean[1024], g_stdev[1024];

    // Iteratively refine the mean and standard deviation
    while (iters <= ITER && fabs(mean - mean_last) > ACC) {
        sum = 0.0f;
        sum_squares = 0.0f;
        count = 0;

        // Reduce data access within loop using a single index calculation
        for (int t = 0; t < nsamp; ++t) {
            float data = (float)d_input[t * nchans + c];
            if (fabs(data - mean) < cutoff ) {
                sum += data;
                sum_squares += data * data;
                count++;
            }
        }
        
        mean_last = mean;
        mean = sum / (float)count;
        sum_squares = sum_squares / count - mean * mean;
        stdev = sqrt(sum_squares);
        cutoff = CUT * stdev;
        iters++;
    }

    // Handle cases of computation exceeding iteration limit or other anomalies
    if (count == 0 || iters > ITER || mean == 0.0f || stdev == 0.0f) {
        for (int t = 0; t < nsamp; t++) {
            d_input[t * nchans + c] = MEAN;
        }
        g_mean[threadIdx.x] = mean = MEAN;
        g_stdev[threadIdx.x] = stdev = 0.0f;
    } else {
        g_mean[threadIdx.x] = mean;
        g_stdev[threadIdx.x] = stdev;
    }

    __syncthreads();

    // Calculate the mean and standard deviation of means and stdevs across block
    float mean_of_mean = 0.0f, stdev_of_mean, mean_of_stdev = 0.0f, stdev_of_stdev;
    sum_squares = 0.0f;
    
    for (int i = 0; i < blockDim.x; i++) {
        mean_of_mean += g_mean[i];
        sum_squares += g_mean[i] * g_mean[i];
    }
    
    mean_of_mean /= blockDim.x;
    sum_squares = sum_squares / blockDim.x - mean_of_mean * mean_of_mean;
    stdev_of_mean = sqrt(sum_squares);

    float m_cutoff = 3.0f * stdev_of_mean;
    sum_squares = 0.0f;
    
    for (int i = 0; i < blockDim.x; i++) {
        mean_of_stdev += g_stdev[i];
        sum_squares += g_stdev[i] * g_stdev[i];
    }

    mean_of_stdev /= blockDim.x;
    sum_squares = sum_squares / blockDim.x - mean_of_stdev * mean_of_stdev;
    stdev_of_stdev = sqrt(sum_squares);

    float s_cutoff = 3.0f * stdev_of_stdev;

    // Filter outliers based on computed cutoffs and normalize data
    if (fabs(g_mean[threadIdx.x] - mean_of_mean) > m_cutoff || fabs(g_stdev[threadIdx.x] - mean_of_stdev) > s_cutoff) {
        for (int t = 0; t < nsamp; t++) {
            d_input[t * nchans + c] = MEAN;
        }
    } else {
        for (int t = 0; t < nsamp; t++) {
            float val = d_input[t * nchans + c];
            d_input[t * nchans + c] = fabs(val - mean) < R_CUT * stdev ? static_cast<unsigned short>(val - mean + MEAN) : MEAN;
        }
    }
}