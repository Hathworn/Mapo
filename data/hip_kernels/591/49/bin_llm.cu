#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bin(unsigned short *d_input, float *d_output, int in_nsamp) {

    // Calculate unique thread ID for flat 2D grid
    int threadIdX = blockIdx.x * blockDim.x + threadIdx.x;
    int threadIdY = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate indexes for input and output
    int out_nsamp = in_nsamp / 2;
    int t_out = threadIdX;
    int t_in = t_out * 2;
    size_t shift_one = threadIdY * out_nsamp + t_out;
    size_t shift_two = threadIdY * in_nsamp + t_in;

    // Perform binning, use shared memory for better performance if feasible
    d_output[shift_one] = (float)(d_input[shift_two] + d_input[shift_two + 1]) / 2.0f;
}