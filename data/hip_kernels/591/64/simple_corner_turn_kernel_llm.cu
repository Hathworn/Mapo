#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_corner_turn_kernel(float *d_input, float *d_output, int primary_size, int secondary_size){
    // Calculate flattened global thread id for optimization
    int idx = blockIdx.y * blockDim.y * primary_size + blockIdx.x * blockDim.x + threadIdx.y * primary_size + threadIdx.x;

    // Check grid boundaries to ensure valid memory access
    if (blockIdx.x * blockDim.x + threadIdx.x < primary_size && blockIdx.y * blockDim.y + threadIdx.y < secondary_size) {
        d_output[blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y + threadIdx.y) * primary_size] = 
            __ldg(&d_input[idx]);
    }
}