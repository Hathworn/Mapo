#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MSD_GPU_Interpolate_linear(float *d_MSD_DIT, float *d_MSD_interpolated, int *d_MSD_DIT_widths, int MSD_DIT_size, int *boxcar, int max_width_performed) {
    int tid = threadIdx.x;
    int desired_width = boxcar[tid];
    
    // Exit early if condition is not met
    if (desired_width > max_width_performed) return;
    
    int position = (int)floorf(log2f((float)desired_width));
    float width1 = d_MSD_DIT_widths[position];
    float mean1 = d_MSD_DIT[position * MSD_RESULTS_SIZE];
    float StDev1 = d_MSD_DIT[position * MSD_RESULTS_SIZE + 1];
    
    if (position == MSD_DIT_size-1 && width1 == (int)desired_width) {
        d_MSD_interpolated[tid * 2] = mean1;
        d_MSD_interpolated[tid * 2 + 1] = StDev1;
    } else {
        // Cache calculations
        float width2 = d_MSD_DIT_widths[position + 1];
        float distance_in_width = width2 - width1;
        float distance_in_mean = d_MSD_DIT[(position + 1) * MSD_RESULTS_SIZE] - mean1;
        float distance_in_StDev = d_MSD_DIT[(position + 1) * MSD_RESULTS_SIZE + 1] - StDev1;

        float width_ratio = ((float)desired_width - width1) / distance_in_width;
        d_MSD_interpolated[tid * 2] = mean1 + distance_in_mean * width_ratio;
        d_MSD_interpolated[tid * 2 + 1] = StDev1 + distance_in_StDev * width_ratio;
    }
}