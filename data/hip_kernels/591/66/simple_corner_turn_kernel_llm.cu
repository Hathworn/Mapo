#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_corner_turn_kernel(unsigned short *d_input, float *d_output, int nchans, int nsamp) {

    // Calculate unique global index for current thread
    size_t threadId = threadIdx.y * blockDim.x + threadIdx.x;
    size_t blockId = blockIdx.y * gridDim.x + blockIdx.x;
    size_t index = blockId * blockDim.x * blockDim.y + threadId;
   
    // Ensure index is within bounds before accessing memory
    if (index < nchans * nsamp) {
        size_t t = index % nsamp;
        size_t c = index / nsamp;
        d_output[c * nsamp + t] = static_cast<float>(__ldg(&d_input[t * nchans + c]));
    }
}