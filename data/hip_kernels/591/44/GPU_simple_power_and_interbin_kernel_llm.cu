#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GPU_simple_power_and_interbin_kernel(float2 *d_input_complex, float *d_output_power, float *d_output_interbinning, int nTimesamples, float norm) {
    int pos_x = blockIdx.x * blockDim.x + threadIdx.x;
    int pos_y = blockIdx.y * ((nTimesamples >> 1) + 1);

    if (pos_x >= (nTimesamples >> 1)) return; // Exit early if outside range

    float2 A = d_input_complex[pos_y + pos_x];
    A.x /= norm; // Normalize A
    A.y /= norm;

    d_output_power[blockIdx.y * (nTimesamples >> 1) + pos_x] = A.x * A.x + A.y * A.y;
    d_output_interbinning[blockIdx.y * nTimesamples + 2 * pos_x] = A.x * A.x + A.y * A.y;

    if (pos_x > 0) {
        float2 B = d_input_complex[pos_y + pos_x + 1];
        B.x /= norm; // Normalize B
        B.y /= norm;

        d_output_interbinning[blockIdx.y * nTimesamples + 2 * pos_x + 1] = 0.616850275f * ((A.x - B.x) * (A.x - B.x) + (A.y - B.y) * (A.y - B.y));
    }
}