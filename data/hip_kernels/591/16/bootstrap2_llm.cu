#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bootstrap2(int bins, int num_els, int num_boots, float *g_idata, double *g_odata, unsigned int *g_irand)
{
    // Cache frequently accessed values
    float myResample = 0.0f;
    int constant = 4294967295 / bins;
    int constant2 = blockIdx.x * bins;
    int dmid = bins * (blockDim.y * blockIdx.y + threadIdx.y);

    // Use shared memory to reduce global memory access
    __shared__ unsigned int s_irand[1024];
    __shared__ float s_idata[1024];
    
    if (threadIdx.x < bins) {
        s_irand[threadIdx.x] = g_irand[constant2 + threadIdx.x];
    }
    
    if (threadIdx.x < bins) {
        s_idata[threadIdx.x] = g_idata[dmid + threadIdx.x];
    }
    
    __syncthreads();

    for (int i = 0; i < bins; i++) {
        int rid = s_irand[i] / constant;
        myResample += s_idata[(rid + threadIdx.x) % bins];
    }

    // Compute output index
    dmid = num_boots * (blockDim.y * blockIdx.y + threadIdx.y);
    g_odata[dmid + threadIdx.x + blockDim.x * blockIdx.x] = ((double)myResample / (double)num_els);
}