#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stretch_kernel(int acc, int samps, float tsamp, float *d_input, float *d_output, float t_zero, float multiplier, float tsamp_inverse) {
    int t = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index

    // Optimize the calculation of p_time and ensure p_time uses only necessary arithmetic
    float p_time = t_zero + multiplier * (t - 1.0f) * t;
    
    int stretch_index = __float2int_rz(p_time * tsamp_inverse); // Efficient conversion of p_time to int

    // Optimize the range check and assignment
    if (stretch_index >= 0 && stretch_index < samps) {
        d_output[stretch_index] = d_input[t];
    }
}