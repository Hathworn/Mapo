#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bootstrap(int bins, int num_els, int num_boots, float *g_idata, double *g_odata, unsigned int *g_irand)
{
    float myResample = 0.0f;

    // Use shared memory to optimize memory access
    extern __shared__ float shared_idata[];
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    
    unsigned int constant = (4294967295 / (bins - blockDim.x));
    int constant2 = blockIdx.x * bins;
    int dmid = bins * (blockDim.y * blockIdx.y + threadIdx.y);

    // Load g_idata into shared memory
    if (tid < bins) {
        shared_idata[tid] = g_idata[tid + dmid];
    }
    __syncthreads();

    for (int i = 0; i < bins; i++) {
        int rid = (g_irand[constant2 + i] / (constant));
        myResample += shared_idata[rid + threadIdx.x];
    }

    dmid = num_boots * (blockDim.y * blockIdx.y + threadIdx.y);
    g_odata[dmid + threadIdx.x + blockDim.x * blockIdx.x] = ((double) myResample / (double) num_els);
}