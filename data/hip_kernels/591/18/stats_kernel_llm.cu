#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stats_kernel(int half_samps, float *d_sum, float *d_sum_square, float *d_signal_power)
{
    int t = blockIdx.x * blockDim.x * STATSLOOP + threadIdx.x;
    
    float sum = 0.0f;
    float sum_square = 0.0f;

    // Use shared memory for thread collaboration
    __shared__ float partial_sum[1024];
    __shared__ float partial_sum_square[1024];

    for (int i = t; i < t + STATSLOOP * blockDim.x; i += blockDim.x)
    {
        float local = d_signal_power[i];
        sum += local;
        sum_square += local * local;
    }

    partial_sum[threadIdx.x] = sum;
    partial_sum_square[threadIdx.x] = sum_square;

    __syncthreads();

    // Reduce sum and sum_square within the block
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        if (threadIdx.x < stride)
        {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + stride];
            partial_sum_square[threadIdx.x] += partial_sum_square[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write the results from the first thread of each block
    if (threadIdx.x == 0)
    {
        d_sum[blockIdx.x] = partial_sum[0];
        d_sum_square[blockIdx.x] = partial_sum_square[0];
    }
}