#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void swap(unsigned short *d_input, float *d_output, int nchans, int nsamp) {
    // Compute the linear index based on block and thread indices
    size_t idx = blockIdx.y * blockDim.y * nsamp + blockIdx.x * blockDim.x + threadIdx.y * nsamp + threadIdx.x;

    // Use shared memory to optimize memory access
    __shared__ float shared_output[1024]; // Assuming blockDim.x * blockDim.y <= 1024

    // Load data into shared memory
    shared_output[threadIdx.y * blockDim.x + threadIdx.x] = __ldg(&d_output[idx]);

    // Ensure all threads have loaded their data into shared memory
    __syncthreads();

    // Perform the write operation
    d_input[idx] = (unsigned short)shared_output[threadIdx.y * blockDim.x + threadIdx.x];
}