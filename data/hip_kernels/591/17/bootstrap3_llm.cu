#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bootstrap3(int bins, int num_els, int num_boots, float *g_idata, double *g_odata, unsigned int *g_irand)
{
    // Use registers to store frequently accessed data
    float myResample = 0.0f;
    int constant = ( 4294967295 / bins );
    int id = threadIdx.x + blockDim.x * blockIdx.x;
    int dmid_base = bins * ( blockDim.y * blockIdx.y + threadIdx.y );
    
    for (int i = 0; i < bins; i++)
    {
        int rid = g_irand[id * bins + i] / constant;
        // Minimize global memory access by using registers
        myResample += g_idata[dmid_base + rid];
    }

    int output_index = num_boots * ( blockDim.y * blockIdx.y + threadIdx.y ) + threadIdx.x + blockDim.x * blockIdx.x;
    g_odata[output_index] = ( (double) myResample / (double) num_els );
}