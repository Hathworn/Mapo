#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rfi_gpu_kernel(unsigned short *d_input, int nchans, int nsamp)
{
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (c >= nchans) return; // Avoids out-of-bounds access

    float stdev = 1000000.0f;
    float mean = 0.0f;
    float cutoff;
    float data;
    float sum, sum_squares;
    int count;

    for(int out = 0; out < 4; out++) {
        sum = 0.0f;
        sum_squares = 0.0f;
        count = 0;
        cutoff = (4.0f * stdev); // Moved inside loop to avoid recalculation

        for(int t = 0; t < nsamp; t++) {
            data = (float)d_input[c * nsamp + t];
            if(data < (mean + cutoff) && data > (mean - cutoff)) {
                sum += data;
                sum_squares += (data * data);
                count++;
            }
        }
        if(count > 0) {
            mean = (sum / (float)count);
            sum_squares = ((sum_squares / count) - (mean * mean));
            stdev = sqrtf(sum_squares); // Use faster sqrtf for single precision
        }
    }

    for(int t = 0; t < nsamp - 4; t++) {
        data = 0.0f;
        for(int x = 0; x < 4; x++) {
            data += (float)d_input[c * nsamp + t + x];
        }
        data *= 0.25f;
        if(data > (mean + cutoff) || data < (mean - cutoff)) {
            d_input[c * nsamp + t] = (unsigned short)mean;
        }
    }
}
```
