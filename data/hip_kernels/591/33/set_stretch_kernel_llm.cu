#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void set_stretch_kernel(int samps, float mean, float *d_input) {
    int t = blockIdx.x * blockDim.x + threadIdx.x;

    // Removed unnecessary check for non-negative index
    if (t < samps) 
        d_input[t] = mean;
}