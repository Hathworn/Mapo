#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PD_ZC_GPU(float *d_input, float *d_output, int maxTaps, int nTimesamples, int nLoops) {
    int x_r, y_r, x_w, y_w;
    int Elements_per_block = PD_NTHREADS * PD_NWINDOWS;
    
    // Precompute commonly used values outside the loop
    y_r = (blockIdx.y * blockDim.y + threadIdx.y) * nTimesamples;
    x_r = (blockIdx.x + 1) * Elements_per_block + threadIdx.x;
    y_w = (blockIdx.y * blockDim.y + threadIdx.y) * (maxTaps - 1) * gridDim.x;
    x_w_init = blockIdx.x * (maxTaps - 1) + threadIdx.x; // Initialize writing offset

    // Unroll loop for better performance if iterations are small
    #pragma unroll
    for (int f = 0; f < nLoops; f++) {
        if (x_r < nTimesamples && threadIdx.x < (maxTaps - 1)) {
            // Compute write index
            int x_w = x_w_init + f * WARP;
            // Perform memory sharing efficiently
            d_output[x_w + y_w] = d_input[x_r + y_r + f * WARP];
        }
    }
}