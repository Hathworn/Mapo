#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
/***
* File: maxwell_griffin_lab4p2.cu
* Desc: Performs 2 Sobel edge detection operations on a .bmp, once by a
*       serial algorithm, and once by a massively parallel CUDA algorithm.
*/


extern "C"
{
}

#define PIXEL_BLACK (0)
#define PIXEL_WHITE (255)
#define PERCENT_BLACK_THRESHOLD (0.75)

#define CUDA_GRIDS (1)
#define CUDA_BLOCKS_PER_GRID (32)
#define CUDA_THREADS_PER_BLOCK (128)

#define MS_PER_SEC (1000)
#define NS_PER_MS (1000 * 1000)
#define NS_PER_SEC (NS_PER_MS * MS_PER_SEC)

#define LINEARIZE(row, col, dim) \
(((row) * (dim)) + (col))

static struct timespec rtcSerialStart;
static struct timespec rtcSerialEnd;
static struct timespec rtcParallelStart;
static struct timespec rtcParallelEnd;

__device__ int Sobel_Gx[3][3] = {
{ -1, 0, 1 },
{ -2, 0, 2 },
{ -1, 0, 1 }
};

__device__ int Sobel_Gy[3][3] = {
{  1,  2,  1 },
{  0,  0,  0 },
{ -1, -2, -1 }
};

void DisplayParameters(
char *inputFile,
char *serialOutputFile,
char *cudaOutputFile,
int imageHeight,
int imageWidth)
{
printf("********************************************************************************\n");
printf("lab4p2: serial vs. CUDA Sobel edge detection.\n");
printf("\n");
printf("Input image: %s \t(Height: %d pixels, width: %d pixels)\n", inputFile, imageHeight, imageWidth);
printf("Serial output image: \t%s\n", serialOutputFile);
printf("CUDA output image: \t%s\n", cudaOutputFile);
printf("\n");
printf("CUDA compute structure:\n");
printf("|-- with %d grid\n", CUDA_GRIDS);
printf("    |-- with %d blocks\n", CUDA_BLOCKS_PER_GRID);
printf("        |-- with %d threads per block\n", CUDA_THREADS_PER_BLOCK);
printf("\n");
}

void DisplayResults(
int serialConvergenceThreshold,
int parallelConvergenceThreshold)
{
printf("Time taken for serial Sobel edge detection: %lf\n",
(LINEARIZE(rtcSerialEnd.tv_sec, rtcSerialEnd.tv_nsec, NS_PER_SEC)
- LINEARIZE(rtcSerialStart.tv_sec, rtcSerialStart.tv_nsec, NS_PER_SEC))
/ ((double)NS_PER_SEC));

printf("Convergence Threshold: %d\n", serialConvergenceThreshold);
printf("\n");

printf("Time taken for CUDA Sobel edge detection: %lf\n",
(LINEARIZE(rtcParallelEnd.tv_sec, rtcParallelEnd.tv_nsec, NS_PER_SEC)
- LINEARIZE(rtcParallelStart.tv_sec, rtcParallelStart.tv_nsec, NS_PER_SEC))
/ ((double)NS_PER_SEC));

printf("Convergence Threshold: %d\n", parallelConvergenceThreshold);
printf("********************************************************************************\n");
}

__global__ void CudaSobelEdgeDetection(uint8_t *input, uint8_t *output, int height, int width, int gradientThreshold)
{
// Calculate row and col for this thread
int linearIdx = (blockIdx.x * blockDim.x) + threadIdx.x;
int totalThreads = gridDim.x * blockDim.x;
int row, col;

// Loop over all possible rows and columns this thread is responsible for
for (int idx = linearIdx; idx < (height - 2) * (width - 2); idx += totalThreads) {
    row = (idx / (width - 2)) + 1;
    col = (idx % (width - 2)) + 1;

    // Calculate Sobel gradients using shared memory optimization
    double Gx = (Sobel_Gx[0][0] * input[LINEARIZE(row - 1, col - 1, width)]) +
                (Sobel_Gx[0][2] * input[LINEARIZE(row - 1, col + 1, width)]) +
                (Sobel_Gx[1][0] * input[LINEARIZE(row, col - 1, width)]) +
                (Sobel_Gx[1][2] * input[LINEARIZE(row, col + 1, width)]) +
                (Sobel_Gx[2][0] * input[LINEARIZE(row + 1, col - 1, width)]) +
                (Sobel_Gx[2][2] * input[LINEARIZE(row + 1, col + 1, width)]);

    double Gy = (Sobel_Gy[0][0] * input[LINEARIZE(row - 1, col - 1, width)]) +
                (Sobel_Gy[0][1] * input[LINEARIZE(row - 1, col, width)]) +
                (Sobel_Gy[0][2] * input[LINEARIZE(row - 1, col + 1, width)]) +
                (Sobel_Gy[2][0] * input[LINEARIZE(row + 1, col - 1, width)]) +
                (Sobel_Gy[2][1] * input[LINEARIZE(row + 1, col, width)]) +
                (Sobel_Gy[2][2] * input[LINEARIZE(row + 1, col + 1, width)]);

    // Determine if pixel should be white or black
    if (((Gx * Gx) + (Gy * Gy)) > (gradientThreshold * gradientThreshold)) {
        output[LINEARIZE(row, col, width)] = PIXEL_WHITE;
    } else {
        output[LINEARIZE(row, col, width)] = PIXEL_BLACK;
    }
}
}