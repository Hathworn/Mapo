#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;

    // Optimize loop unrolling for better performance
    while (eidx < numElements) {
        shmem[threadIdx.x] += a[eidx] * b[eidx];
        eidx += gridDim.x * DP_BLOCKSIZE;
    }

    __syncthreads();

    // Unroll the reduction loop using warp-level primitives
    if (threadIdx.x < 32) {
        for (int offset = 16; offset > 0; offset /= 2) {
            shmem[threadIdx.x] += __shfl_down_sync(0xffffffff, shmem[threadIdx.x], offset);
        }
    }

    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}