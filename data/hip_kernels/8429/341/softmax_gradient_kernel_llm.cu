#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    float local_sum = 0.0f;

    // Efficient reduction using warp-level operations
    for (int i = idx; i < dim; i += blockDim.x) {
        local_sum += dY[i] * Y[i];
    }
    reduction_buffer[idx] = local_sum;
    __syncthreads();

    // Use a single warp to complete reduction
    if (idx < 32) {
        float warp_sum = reduction_buffer[idx];
        for (int offset = 32; offset < blockDim.x; offset += 32) {
            warp_sum += reduction_buffer[offset + idx];
        }
        reduction_buffer[idx] = warp_sum;
    }
    __syncthreads();

    // Compute gradient
    float grad_sum = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - grad_sum);
    }
}