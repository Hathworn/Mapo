#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Use `__syncthreads()` to ensure all threads have initialized before usage
    __shared__ hiprandState localState[NUM_RND_THREADS_PER_BLOCK];
    hiprand_init(seed, tidx, 0, &localState[threadIdx.x]);
    __syncthreads();
    state[tidx] = localState[threadIdx.x];
}