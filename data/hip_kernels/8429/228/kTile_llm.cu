#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    
    // Prefetch frequently used values to avoid redundant computations
    const uint srcWidthTimesSrcHeight = srcWidth * srcHeight;
    
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        
        // Modulo is computationally expensive; optimize it if necessary
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        
        // Load source element with computed indices
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}