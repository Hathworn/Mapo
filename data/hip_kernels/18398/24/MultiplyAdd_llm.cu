#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate 2D index using block and thread indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate 1D index efficiently
    int p = y * width + x;

    // Perform operation only if indices are within bounds
    if (x < width && y < height)
    {
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
}