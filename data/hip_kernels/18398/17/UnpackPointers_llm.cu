#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;

    // Optimize by calculating y and x increment more effectively
    for (int y = 0; y < h; y += 32) {
        for (int x = tx; x < w; x += 16) {
            unsigned int val = minmax[y * w + x];
            if (val) {
                int baseIdx = y * w + x;  // Base index calculation outside loop
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = baseIdx + k * w;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}