#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];

    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    const int b = blockDim.x;

    int p = y * width + x;

    // Initialize shared memory with conditionally assigned values
    float val = (x < width) ? d_Data[p] : d_Data[p - x];
    minvals[tx] = val;
    maxvals[tx] = val;

    // Process remaining rows in the block
    for (int ty = 1; ty < 16; ty++) {
        p += width;
        if (x < width) {
            val = d_Data[p];
            // Update min and max values
            if (val < minvals[tx]) minvals[tx] = val;
            if (val > maxvals[tx]) maxvals[tx] = val;
        }
    }
    __syncthreads();

    // Reduce min and max values within the block
    int mod = 1;
    for (int d = 1; d < b; d <<= 1) {
        if ((tx & mod) == 0) {
            float minCandidate = minvals[tx + d];
            float maxCandidate = maxvals[tx + d];
            if (minCandidate < minvals[tx]) minvals[tx] = minCandidate;
            if (maxCandidate > maxvals[tx]) maxvals[tx] = maxCandidate;
        }
        mod = (mod << 1) | 1;
        __syncthreads();
    }

    // Write the block's min and max to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}