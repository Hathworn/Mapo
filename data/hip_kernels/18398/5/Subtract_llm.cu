#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    // Calculate global thread indices
    const int x = blockIdx.x * SUBTRACT_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACT_H + threadIdx.y;

    // Compute linear index within the 2D grid
    int p = y * pitch + x;

    // Perform subtraction only if within bounds
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}