#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    // Calculate the 3D thread index
    const int x = blockIdx.x * SUBTRACTM_W + threadIdx.x;
    const int y = blockIdx.y * SUBTRACTM_H + threadIdx.y;
    const int z = threadIdx.z;
    
    // Use a single calculation for size
    const int sz = height * pitch;
    const int p = z * sz + y * pitch + x;
    
    // Check boundaries before calculation
    if (x < width && y < height)
    {
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
    
    // __syncthreads is unnecessary here as there are no shared memory dependencies
}