#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Calculate global thread coordinates without deprecated __mul24.
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate index and ensure thread is within bounds.
    int p = y * width + x;
    if (x < width && y < height)
    {
        // Perform subtraction only if within bounds.
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // Remove __syncthreads() as it's unnecessary here.
}