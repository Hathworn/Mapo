#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int height)
{
    // Data cache
    __shared__ float data1[3*(MINMAX_SIZE + 2)];
    __shared__ float data2[3*(MINMAX_SIZE + 2)];
    __shared__ float data3[3*(MINMAX_SIZE + 2)];
    __shared__ float ymin1[(MINMAX_SIZE + 2)];
    __shared__ float ymin2[(MINMAX_SIZE + 2)];
    __shared__ float ymin3[(MINMAX_SIZE + 2)];
    __shared__ float ymax1[(MINMAX_SIZE + 2)];
    __shared__ float ymax2[(MINMAX_SIZE + 2)];
    __shared__ float ymax3[(MINMAX_SIZE + 2)];

    // Current tile and apron limits, relative to row start
    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_SIZE;
    const int xEnd = xStart + MINMAX_SIZE - 1;
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;
    const int xEndClamped = min(xEnd, width - 1);
    const int memWid = MINMAX_SIZE + 2;

    int memPos0 = (tx - WARP_SIZE + 1);
    int memPos1 = memPos0;
    int yq = 0;
    unsigned int output = 0;
    for (int y = 0; y < 34; y++) {
        output >>= 1;
        int memPos = yq * memWid + memPos0;
        int yp = blockIdx.y * 32 + y - 1;
        yp = clamp(yp, 0, height - 1);
        int readStart = yp * width;

        // Set the entire data cache contents
        if (tx >= (WARP_SIZE - 1)) {
            float3 dataRead;
            if (xReadPos >= 0 && xReadPos < width) {
                dataRead = make_float3(d_Data1[readStart + xReadPos], d_Data2[readStart + xReadPos], d_Data3[readStart + xReadPos]);
            } else {
                dataRead = make_float3(0, 0, 0); // Load zeros for out-of-bounds
            }
            data1[memPos] = dataRead.x;
            data2[memPos] = dataRead.y;
            data3[memPos] = dataRead.z;
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;
        if (y > 1) {
            if (tx < memWid) {
                float3 minValues = make_float3(fminf(fminf(data1[memPos0], data1[memPos1]), data1[memPos2]),
                                               fminf(fminf(data2[memPos0], data2[memPos1]), data2[memPos2]),
                                               fminf(fminf(data3[memPos0], data3[memPos1]), data3[memPos2]));
                float3 maxValues = make_float3(fmaxf(fmaxf(data1[memPos0], data1[memPos1]), data1[memPos2]),
                                               fmaxf(fmaxf(data2[memPos0], data2[memPos1]), data2[memPos2]),
                                               fmaxf(fmaxf(data3[memPos0], data3[memPos1]), data3[memPos2]));

                ymin1[tx] = minValues.x;
                ymin2[tx] = fminf(fminf(minValues.x, minValues.y), minValues.z);
                ymin3[tx] = minValues.z;
                ymax1[tx] = maxValues.x;
                ymax2[tx] = fmaxf(fmaxf(maxValues.x, maxValues.y), maxValues.z);
                ymax3[tx] = maxValues.z;
            }
        }
        __syncthreads();

        if (y > 1) {
            if (tx < MINMAX_SIZE && xWritePos <= xEndClamped) {
                float middleValue = data2[memPos1 + 1];
                float minv = fminf(fminf(fminf(fminf(ymin2[tx], ymin2[tx + 2]), ymin1[tx + 1]), ymin3[tx + 1]), middleValue);
                minv = fminf(minv, d_ConstantA[1]);
                float maxv = fmaxf(fmaxf(fmaxf(fmaxf(ymax2[tx], ymax2[tx + 2]), ymax1[tx + 1]), ymax3[tx + 1]), middleValue);
                maxv = fmaxf(maxv, d_ConstantA[0]);
                if (middleValue < minv || middleValue > maxv)
                    output |= 0x80000000;
            }
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }
    if (tx < MINMAX_SIZE && xWritePos < width) {
        int writeStart = blockIdx.y * width + xWritePos;
        d_Result[writeStart] = output;
    }
}