#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts) {
    const int tx = threadIdx.x;
    int numPts = 0;
    // Calculate the number of rows to process per block for coalesced memory access
    int rowsPerBlock = h / 32;
    int tidx = blockIdx.x * blockDim.x + tx;

    // Loop over calculated rows and iterate only if within range
    for (int y = 0; y < rowsPerBlock && numPts < maxPts; y++) {
        for (int x = 0; x < w; x += blockDim.x) {
            unsigned int val = minmax[y * w + x + tx];
            if (val) {
                for (int k = 0; val != 0 && k < 32 && numPts < maxPts; k++) {
                    if (val & 0x1) {
                        if (numPts < maxPts) {
                            ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                            numPts++;
                        }
                    }
                    val >>= 1;
                }
            }
        }
    }
}