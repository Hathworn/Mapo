#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ static void myAtomicAdd(float *address, float value)
{
#if __CUDA_ARCH__ >= 200
    atomicAdd(address, value);
#else
    // cf. https://www.sharcnet.ca/help/index.php/CUDA_tips_and_tricks
    int oldval, newval, readback;

    oldval = __float_as_int(*address);
    newval = __float_as_int(__int_as_float(oldval) + value);
    while ((readback=atomicCAS((int *)address, oldval, newval)) != oldval)
    {
        oldval = readback;
        newval = __float_as_int(__int_as_float(oldval) + value);
    }
#endif
}

__global__ void computeCSRColSums(float *d_colSums, const float *d_systemMatrixVals, const int *d_systemMatrixRows, const int *d_systemMatrixCols, const size_t m, const size_t n)
{
    const size_t row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row >= m)
        return;
        
    // Load row starts and ends in shared memory for coalesced access
    __shared__ int sharedRows[32]; // Assuming blockDim.x <= 32

    if (threadIdx.x == 0 || threadIdx.x == blockDim.x) {
        if (row <= m)
            sharedRows[threadIdx.x] = d_systemMatrixRows[row];
        if (row + 1 <= m)
            sharedRows[threadIdx.x + 1] = d_systemMatrixRows[row + 1];
    }
    __syncthreads();

    // Iterate over the elements within the row
    int rowStart = sharedRows[threadIdx.x];
    int rowEnd = sharedRows[threadIdx.x + 1];

    for (int cidx = rowStart; cidx < rowEnd; ++cidx)
    {
        myAtomicAdd(d_colSums + d_systemMatrixCols[cidx], d_systemMatrixVals[cidx]);
    }
}