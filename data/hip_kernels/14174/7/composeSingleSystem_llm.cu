#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int roundToInt(float val)
{
    return (int)floor(val + 0.5f);
}
__device__ float d_priorF;  

__global__ void add(float *p, float *q) { *p += *q; }

__global__ void composeSingleSystem(const size_t offset, const float *H, const size_t lowresWidth, const size_t lowresHeight, 
                                    const size_t highresWidth, const size_t highresHeight, const float psfWidth, 
                                    const int pixelRadius, float *systemMatrixVals, int *systemMatrixCols, int *systemMatrixRows)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t lowresPixels  = lowresWidth  * lowresHeight;

    if (idx >= lowresPixels) return;

    size_t x = idx % lowresWidth;
    size_t y = idx / lowresWidth;
    size_t r = idx + offset;

    // Precompute constants outside loops to avoid repeated calculations
    float zoom = float(highresWidth) / float(lowresWidth);
    float zoomSquared = zoom * zoom;
    float invPSFWidthSquared = 1.0f / (2.0f * zoomSquared * psfWidth * psfWidth);

    float hrx = (H[0] * x + H[1] * y + H[2]) * zoom;
    float hry = (H[3] * x + H[4] * y + H[5]) * zoom;

    float weightSum = 0.0f;
    const size_t maxRowElems = (2 * pixelRadius + 1) * (2 * pixelRadius + 1);
    size_t offsetCRS = 0;
    size_t offsetRows = maxRowElems * r;

    for (int offsetY = -pixelRadius; offsetY <= pixelRadius; ++offsetY)
    {
        int ny = roundToInt(hry + offsetY);
        if (ny < 0 || ny >= highresHeight) continue;

        for (int offsetX = -pixelRadius; offsetX <= pixelRadius; ++offsetX)
        {
            int nx = roundToInt(hrx + offsetX);
            if (nx < 0 || nx >= highresWidth) continue;

            float dx = hrx - nx;
            float dy = hry - ny;

            // Combine repeated terms
            float dist = (dx * dx * H[0] * H[0] + dy * dy * H[4] * H[4] + dx * dy * (H[0] * H[3] + H[1] * H[4]));
            float weight = expf(-dist * invPSFWidthSquared);

            size_t valIdx = offsetRows + offsetCRS;
            systemMatrixVals[valIdx] = weight;
            systemMatrixCols[valIdx] = ny * highresWidth + nx;

            weightSum += weight;
            ++offsetCRS;
        }
    }

    if (weightSum > 0.0f)
    {
        float invWeightSum = 1.0f / weightSum;
        #pragma unroll // Unroll loop to improve performance
        for (size_t i = 0; i < offsetCRS; ++i)
        {
            systemMatrixVals[offsetRows + i] *= invWeightSum;
        }
    }

    if (offsetCRS == 0)
    {
        systemMatrixVals[offsetRows] = 0.0f;
        systemMatrixCols[offsetRows] = 0;
        ++offsetCRS;
    }

    bool copy = false;
    while (offsetCRS < maxRowElems)
    {
        const size_t idx = offsetRows + offsetCRS;
        if (systemMatrixCols[idx - 1] + 1 >= highresWidth * highresHeight)
        {
            copy = true;
            break;
        }
        systemMatrixVals[idx] = 0.0f;
        systemMatrixCols[idx] = systemMatrixCols[idx - 1] + 1;
        offsetCRS++;
    }

    if (copy)
    {
        for (int idx = offsetCRS - 1; idx >= 0; --idx)
        {
            systemMatrixVals[offsetRows + maxRowElems - (offsetCRS - idx)] =
                systemMatrixVals[offsetRows + idx];
            systemMatrixCols[offsetRows + maxRowElems - (offsetCRS - idx)] =
                systemMatrixCols[offsetRows + idx];
        }

        for (int idx = maxRowElems - offsetCRS - 1; idx >= 0; --idx)
        {
            systemMatrixVals[offsetRows + idx] = 0.0f;
            systemMatrixCols[offsetRows + idx] =
                systemMatrixCols[offsetRows + idx + 1] - 1;
        }
    }

    systemMatrixRows[r] = r * maxRowElems;
}