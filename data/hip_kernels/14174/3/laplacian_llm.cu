#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void laplacian(float *dst, const float *src, const size_t width, const size_t height, const size_t pixelsPerThread)
{
    // Calculate global index
    const size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t col = index % width;
    const size_t crow = index / width * pixelsPerThread;

    // Boundary check
    if (col >= width || crow >= height)
        return;

    // Calculate start and end row for the current thread
    const size_t srow = crow + 1;
    const size_t erow = min(crow + pixelsPerThread - 1, height - 1);

    for (size_t row = crow; row <= erow; ++row)
    {
        const size_t idx = row * width + col;
        
        // Initialize with central pixel
        float value = src[idx] * 1.0f;

        // Subtract neighbors
        if (row + 1 < height) value -= 0.25f * src[idx + width]; // S
        if (row > 0)          value -= 0.25f * src[idx - width]; // N
        if (col + 1 < width)  value -= 0.25f * src[idx + 1];     // E
        if (col > 0)          value -= 0.25f * src[idx - 1];     // W

        // Store resulting value
        dst[idx] = value;
    }
}