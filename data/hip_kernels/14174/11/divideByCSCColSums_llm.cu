#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void divideByCSCColSums(const float *values, const int *colPointers, float *pixels, const size_t n)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;

    // Use shared memory to minimize global memory access
    __shared__ float sharedWeight;

    // Each thread calculates its weight
    float weight = 0.0f;
    for (size_t ridx = colPointers[idx]; ridx < colPointers[idx + 1]; ++ridx)
    {
        weight += values[ridx];
    }

    // Store weight in shared memory for subsequent access
    sharedWeight = weight;
    __syncthreads();

    // Divide pixel value by the accumulated weight in shared memory
    pixels[idx] /= sharedWeight + 1e-6f;
}