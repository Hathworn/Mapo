#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void huber(float *a, const size_t width, const size_t height, const float alpha, const float strength, const size_t pixelsPerThread, float *f)
{
    const size_t globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t col = globalIdx % width;
    const size_t crow = globalIdx / width * pixelsPerThread;
    
    // Return if out of bounds
    if (col >= width || crow >= height)
        return;

    const size_t erow = min(static_cast<unsigned int>(crow + pixelsPerThread), static_cast<unsigned int>(height));
    const float alpha2 = alpha * alpha;
    float colF = 0.0f;

    // Loop through rows assigned to this thread
    for (size_t row = crow; row < erow; ++row)
    {
        const size_t idx = row * width + col;
        
        // Optimized Pseudo-Huber loss function computation
        const float value = a[idx];
        const float root = sqrtf(1.0f + value * value / alpha2);
        const float increment = alpha2 * (root - 1.0f);
        
        colF += increment;
        a[idx] *= strength / root;
    }

    colF *= strength;
    
    // Store result back to global memory
    f[globalIdx] = colF;
}