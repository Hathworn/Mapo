#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwiseDiv(float *a, const float *b, const size_t len)
{
    // Calculate index
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop to handle array elements beyond blockDim.x
    for (; idx < len; idx += gridDim.x * blockDim.x) {
        a[idx] /= b[idx] + 1e-6f;
    }
}