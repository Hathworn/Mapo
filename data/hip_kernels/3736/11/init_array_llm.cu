#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_array(int *g_data, int *factor, int num_iterations)
{
    // Use shared memory for better performance
    __shared__ int shared_factor;
    if (threadIdx.x == 0) {
        shared_factor = *factor;
    }
    __syncthreads();

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = 0; i < num_iterations; i++) {
        g_data[idx] += shared_factor; // Use shared memory
    }
}