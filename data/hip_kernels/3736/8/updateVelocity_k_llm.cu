#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateVelocity_k(float2 *v, float *vx, float *vy, int dx, int pdx, int dy, int lb, size_t pitch) {
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * blockDim.y * lb + threadIdx.y * lb;

    // Ensure threads are within the domain boundaries
    if (gtidx < dx) {
        float scale = 1.f / (dx * dy);

        for (int p = 0; p < lb; p++) {
            int fi = gtidy + p;
            if (fi < dy) {
                int fjr = fi * pdx + gtidx;

                // Utilize shared memory if beneficial (not implemented here, depends on memory access pattern)
                
                // Load and normalize velocity components
                float vxterm = vx[fjr];
                float vyterm = vy[fjr];

                float2 nvterm;
                nvterm.x = vxterm * scale;
                nvterm.y = vyterm * scale;

                // Use pointer offset technique for memory efficiency
                float2 *fj = (float2*)((char*)v + fi * pitch) + gtidx;
                *fj = nvterm;
            }
        }
    }
}