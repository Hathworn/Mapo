#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void advectParticles_OGL(float2 *part, float2 *v, int dx, int dy, float dt, int lb, size_t pitch) {

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;

    if (gtidx < dx) {
        for (int p = 0; p < lb; p++) {
            int fi = gtidy + p;
            if (fi < dy) {
                int fj = fi * dx + gtidx;
                float2 pterm = part[fj];
                
                // Use clamping for boundary conditions
                int xvi = min(max((int)(pterm.x * dx), 0), dx - 1);
                int yvi = min(max((int)(pterm.y * dy), 0), dy - 1);
                
                // Load velocity using faster memory access pattern
                float2 vterm = *((float2*)((char*)v + yvi * pitch) + xvi);

                // Optimize position updates using fmodf for wrapping
                pterm.x = fmodf(pterm.x + dt * vterm.x + 1.0f, 1.0f);
                pterm.y = fmodf(pterm.y + dt * vterm.y + 1.0f, 1.0f);

                part[fj] = pterm;
            }
        }
    }
}