#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void square(int *array, int arrayCount)
{
    // Use shared memory to optimize
    extern __shared__ int dynamicSmem[];
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < arrayCount) {
        dynamicSmem[threadIdx.x] = array[idx];
        __syncthreads(); // Ensure all threads have loaded data to shared memory

        array[idx] = dynamicSmem[threadIdx.x] * dynamicSmem[threadIdx.x];
    }
}