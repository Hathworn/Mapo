#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "addForces_k_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float2 *v = NULL;
hipMalloc(&v, XSIZE*YSIZE);
int dx = 1;
int dy = 1;
int spx = 1;
int spy = 1;
float fx = 1;
float fy = 1;
int r = 1;
size_t pitch = 2;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
addForces_k<<<gridBlock, threadBlock>>>(v,dx,dy,spx,spy,fx,fy,r,pitch);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
addForces_k<<<gridBlock, threadBlock>>>(v,dx,dy,spx,spy,fx,fy,r,pitch);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
addForces_k<<<gridBlock, threadBlock>>>(v,dx,dy,spx,spy,fx,fy,r,pitch);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}