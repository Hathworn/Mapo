#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addForces_k(float2 *v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch) {

    // Calculate thread-specific indices
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    // Optimize memory access by calculating pointer position once
    float2 *fj = (float2*)((char*)v + (ty + spy) * pitch) + tx + spx;
    
    // Load data into a register to reduce memory latency
    float2 vterm = *fj;

    // Pre-calculate terms for improved performance
    int tx_shifted = tx - r;
    int ty_shifted = ty - r;
    float s = 1.f / (1.f + tx_shifted * tx_shifted * tx_shifted * tx_shifted + 
                          ty_shifted * ty_shifted * ty_shifted * ty_shifted);

    // Perform computation using registers
    vterm.x += s * fx;
    vterm.y += s * fy;

    // Store result back to the memory
    *fj = vterm;
}