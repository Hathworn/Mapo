#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void diffuseProject_k(float2 *vx, float2 *vy, int dx, int dy, float dt, float visc, int lb) {
    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;

    // Load shared memory for vx and vy to improve memory access patterns
    extern __shared__ float2 sharedMem[];
    float2 *shared_vx = sharedMem;
    float2 *shared_vy = sharedMem + dx * lb * blockDim.y;

    if (gtidx < dx) {
        for (int p = 0; p < lb; p++) {
            int fi = gtidy + p;
            if (fi < dy) {
                int fj = fi * dx + gtidx;
                
                // Load data into shared memory
                shared_vx[threadIdx.y * lb + p] = vx[fj];
                shared_vy[threadIdx.y * lb + p] = vy[fj];

                __syncthreads(); // Ensure all data is loaded before processing

                float2 xterm = shared_vx[threadIdx.y * lb + p];
                float2 yterm = shared_vy[threadIdx.y * lb + p];

                int iix = gtidx;
                int iiy = (fi > dy / 2) ? (fi - dy) : fi;

                float kk = (float)(iix * iix + iiy * iiy);
                float diff = 1.f / (1.f + visc * dt * kk);
                xterm.x *= diff;
                xterm.y *= diff;
                yterm.x *= diff;
                yterm.y *= diff;

                if (kk > 0.f) {
                    float rkk = 1.f / kk;
                    float rkp = (iix * xterm.x + iiy * yterm.x);
                    float ikp = (iix * xterm.y + iiy * yterm.y);
                    xterm.x -= rkk * rkp * iix;
                    xterm.y -= rkk * ikp * iix;
                    yterm.x -= rkk * rkp * iiy;
                    yterm.y -= rkk * ikp * iiy;
                }

                // Store result from shared memory back to global memory
                vx[fj] = xterm;
                vy[fj] = yterm;
            }
            __syncthreads(); // Ensure all changes are done before next iteration
        }
    }
}