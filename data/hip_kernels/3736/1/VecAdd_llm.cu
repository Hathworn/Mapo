#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAdd(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x; // Calculate global thread index
    if (i < N) 
    {
        C[i] = A[i] + B[i]; // Perform vector addition only if index is within bounds
    }
}