#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarySearch(const int limit, const int databaseSize, const long* databaseArray, const long* inputArray, int* outputArray) {
    // Use shared memory for improved memory access efficiency
    __shared__ long sharedDatabaseArray[1024]; // Assuming databaseSize <= 1024 for simplicity
    const int bIdx = gridDim.x * blockIdx.y + blockIdx.x;
    const int tIdx = blockDim.x * bIdx + threadIdx.x;

    // Load part of databaseArray into shared memory
    for (int i = threadIdx.x; i < databaseSize; i += blockDim.x) {
        sharedDatabaseArray[i] = databaseArray[i];
    }
    __syncthreads(); // Ensure shared memory is fully loaded

    if (tIdx < limit) {
        const long input = inputArray[tIdx];
        int output = -1;

        int startIdx = 0;
        int endIdx = databaseSize - 1;
        while (startIdx <= endIdx) {
            const int idx = (startIdx + endIdx) >> 1;
            const long temp = sharedDatabaseArray[idx]; // Use shared memory
            if (temp < input) {
                startIdx = idx + 1;
            } else if (temp == input) {
                output = idx;
                break;
            } else {
                endIdx = idx - 1;
            }
        }

        outputArray[tIdx] = output;
    }
}