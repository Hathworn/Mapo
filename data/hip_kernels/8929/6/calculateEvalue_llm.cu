#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculateEvalue(const int q_begin, const int matchSize, const double totalDatabaseSize, 
                                const double K, const double lambda, const int* queryLengthArray, 
                                const int* queryIDArray, const int* scoreArray, double* evalueArray) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimize memory accesses by reducing index calculations
    if (idx < matchSize) {
        int queryID = queryIDArray[idx] - q_begin; // Cache value to avoid repeated access
        int queryLength = queryLengthArray[queryID];
        int score = scoreArray[idx];

        // Perform computation and store result
        evalueArray[idx] = K * totalDatabaseSize * queryLength * exp(-lambda * score);
    }
}