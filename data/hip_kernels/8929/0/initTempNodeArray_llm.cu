#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initTempNodeArray(const int hitNum, const int allowableGap, int* tempNodeArray_score, int* tempNodeArray_vertical, int* tempNodeArray_horizontal, int* tempNodeArray_matchNum) {
    // Calculate unique global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    const int halfTempNodeWidth = allowableGap + MARGIN;
    const int tempNodeWidth = 1 + 2 * halfTempNodeWidth;
    
    // Loop unrolling for efficiency, combined if condition
    if (idx < hitNum * tempNodeWidth) {
        const int bandIdx = idx / hitNum;
        
        int score, vertical, horizontal, matchNum;
        
        if (bandIdx < halfTempNodeWidth) {
            score = vertical = horizontal = matchNum = -30000;
        } else if (bandIdx == halfTempNodeWidth) {
            score = matchNum = 0;
            vertical = horizontal = GAP_OPEN_POINT;
        } else {
            const int i = bandIdx - halfTempNodeWidth;
            const int tempScore = i * GAP_POINT + GAP_OPEN_POINT;
            score = tempScore;
            vertical = tempScore + GAP_OPEN_POINT;
            horizontal = tempScore;
            matchNum = 0;
        }
        
        tempNodeArray_score[idx] = score;
        tempNodeArray_vertical[idx] = vertical;
        tempNodeArray_horizontal[idx] = horizontal;
        tempNodeArray_matchNum[idx] = matchNum;
    }
}