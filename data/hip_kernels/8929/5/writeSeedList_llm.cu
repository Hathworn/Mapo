#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeSeedList(const int idxLimit, const int* __restrict__ gatewayIndexArray, const int* __restrict__ indexArray, const int* __restrict__ seedWriteIndexArray, const int* __restrict__ cellSizeArray, const int* __restrict__ tIDArray, const int* __restrict__ tIndexArray, const int* __restrict__ qIDArray, const int* __restrict__ qIndexArray, int* __restrict__ target_IDArray, int* __restrict__ target_indexArray, int* __restrict__ query_IDArray, int* __restrict__ query_indexArray) {
    const int tIdx = blockDim.x * (blockIdx.y * gridDim.x + blockIdx.x) + threadIdx.x;
    if (tIdx < idxLimit) {
        const int qID = qIDArray[tIdx];
        const int qIdx = qIndexArray[tIdx];
        
        const int seedWriteIndex = seedWriteIndexArray[tIdx];
        const int cellSize = cellSizeArray[tIdx];
        const int gatewayIndex = gatewayIndexArray[tIdx];

        #pragma unroll // Unroll loop for performance
        for (int i = 0; i < cellSize; ++i) {
            const int seedListIndex = seedWriteIndex + i;
            const int refTargetIndex = indexArray[gatewayIndex + i];
            target_IDArray[seedListIndex] = tIDArray[refTargetIndex];
            target_indexArray[seedListIndex] = tIndexArray[refTargetIndex];
            query_IDArray[seedListIndex] = qID;
            query_indexArray[seedListIndex] = qIdx;
        }
    }
}
```
