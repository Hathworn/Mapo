#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate the global thread index once to avoid recomputation
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Each thread gets the same seed with different sequence number and no offset
    hiprand_init(seed, tidx, 0, &state[tidx]);
}