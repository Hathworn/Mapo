#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    float thread_sum = 0.0f;
    // Use unrolled loop for better performance.
    for (int i = idx; i < dim; i += blockDim.x) {
        thread_sum += dY[i] * Y[i];
    }
    reduction_buffer[idx] = thread_sum;
    __syncthreads();

    // Optimize reduction with parallel tree reduction.
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    float inner_product = reduction_buffer[0];

    // Compute gradient.
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - inner_product);
    }
}