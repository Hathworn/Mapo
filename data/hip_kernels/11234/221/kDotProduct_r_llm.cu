#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;
    
    // Ensure that the loop boundary checks with actual numElements
    if (eidx < numElements) {
        // Unroll loop for better performance
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            shmem[threadIdx.x] += a[eidx] * b[eidx];
        }
    }
    __syncthreads();

    // Optimize reduction using a single warp for final steps
    if (threadIdx.x < 256) {
        shmem[threadIdx.x] += shmem[threadIdx.x + 256];
    }
    __syncthreads();
    if (threadIdx.x < 128) {
        shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    }
    __syncthreads();
    if (threadIdx.x < 64) {
        shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    }
    __syncthreads();

    // Use warp shuffle operations for faster reduction
    if (threadIdx.x < 32) {
        volatile float* mysh = &shmem[threadIdx.x];
        *mysh += mysh[32];
        *mysh += mysh[16];
        *mysh += mysh[8];
        *mysh += mysh[4];
        *mysh += mysh[2];
        *mysh += mysh[1];
    }

    // Write result from the first thread of the block
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}