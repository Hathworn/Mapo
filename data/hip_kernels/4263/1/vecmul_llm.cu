#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecmul(float *A, float* B, float *C, int size)
{
    // Optimized: cache for better memory bandwidth usage
    __shared__ float sharedA[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float sharedB[BLOCK_SIZE][BLOCK_SIZE];

    // Calculate block row and column
    int blockRow = blockIdx.y * blockDim.y;
    int blockCol = blockIdx.x * blockDim.x;

    // Row and Column indexes:
    int row = blockRow + threadIdx.y;
    int col = blockCol + threadIdx.x;

    float result = 0;

    // Iterate over tiles
    for (int tileIdx = 0; tileIdx < size; tileIdx += BLOCK_SIZE) {

        // Load elements into shared memory
        if (row < size && (tileIdx + threadIdx.x) < size) {
            sharedA[threadIdx.y][threadIdx.x] = A[row * size + tileIdx + threadIdx.x];
        } else {
            sharedA[threadIdx.y][threadIdx.x] = 0.0;
        }

        if ((tileIdx + threadIdx.y) < size && col < size) {
            sharedB[threadIdx.y][threadIdx.x] = B[(tileIdx + threadIdx.y) * size + col];
        } else {
            sharedB[threadIdx.y][threadIdx.x] = 0.0;
        }

        // Synchronize before computation
        __syncthreads();

        // Compute partial result
        for (int ix = 0; ix < BLOCK_SIZE; ix++) {
            result += sharedA[threadIdx.y][ix] * sharedB[ix][threadIdx.x];
        }

        // Synchronize again before next tile
        __syncthreads();
    }

    // Store the result
    if (row < size && col < size) {
        C[row * size + col] = result;
    }
}