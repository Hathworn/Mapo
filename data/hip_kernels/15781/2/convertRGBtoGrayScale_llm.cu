#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convertRGBtoGrayScale(uint8_t* src, uint8_t* dst, int width, int height, int channels)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Ensure thread safety and bounds checking
    if (x < width && y < height) {
        int grayOffset = y * width + x;
        int rgbOffset = grayOffset * channels;
        
        unsigned char r = src[rgbOffset];          // red value for pixel
        unsigned char g = src[rgbOffset + 1];      // corrected index for green
        unsigned char b = src[rgbOffset + 2];      // corrected index for blue
        
        // Calculate grayscale value using linear approximation
        dst[grayOffset] = __float2uint_rn(0.21f * r + 0.71f * g + 0.07f * b);
    }
}