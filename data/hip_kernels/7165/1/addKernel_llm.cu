#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use blockIdx and blockDim for scalable parallelism
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure memory access is within bounds (assuming n is size of arrays)
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}