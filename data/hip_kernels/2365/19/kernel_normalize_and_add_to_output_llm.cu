#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Macro for fast integer multiplication
#define __iMul24(a, b) ((a) * (b))

// Optimized kernel function
__global__ void kernel_normalize_and_add_to_output(float * dev_vol_in, float * dev_vol_out, float * dev_accumulate_weights, float * dev_accumulate_values)
{
    // Use the recommended method for computing unique thread index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Early exit if indices are out of bounds
    if (i >= c_volSize.x || j >= c_volSize.y || k >= c_volSize.z) return;

    // Index row major into the volume
    long int out_idx = i + (j + k * c_volSize.y) * c_volSize.x;
    float eps = 1e-6f;

    // Efficient conditional operation for normalization
    float weight = dev_accumulate_weights[out_idx];
    float acc_value = dev_accumulate_values[out_idx];

    if (c_normalize)
    {
        if (fabsf(weight) > eps) // Use more efficient fabsf for float
        {
            dev_vol_out[out_idx] = dev_vol_in[out_idx] + (acc_value / weight);
        }
        else
        {
            dev_vol_out[out_idx] = dev_vol_in[out_idx];
        }
    }
    else
    {
        dev_vol_out[out_idx] = dev_vol_in[out_idx] + acc_value;
    }
}