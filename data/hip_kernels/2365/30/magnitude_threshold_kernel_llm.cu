#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void magnitude_threshold_kernel(float * grad_x, float * grad_y, float * grad_z, float gamma)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    // Ensure index is within bounds
    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
        return;

    // Compute unique ID using 1D indexing
    long int id = (k * c_Size.y + j) * c_Size.x + i;

    // Use fast math operations for computing norm
    float norm = __fsqrt_rn(grad_x[id] * grad_x[id] + grad_y[id] * grad_y[id] + grad_z[id] * grad_z[id]);
    
    if (norm > gamma)
    {
        float ratio = gamma / norm;
        // Apply ratio using in-place operations
        grad_x[id] *= ratio;
        grad_y[id] *= ratio;
        grad_z[id] *= ratio;
    }
}