#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void multiply_by_beta_kernel(float *input, float *output, float beta)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idz = blockIdx.z * blockDim.z + threadIdx.z;

    // Calculate unique identifier for the thread
    long int id = (idz * c_Size.y + idy) * c_Size.x + idx;

    // Check indices and perform multiplication
    if (idx < c_Size.x && idy < c_Size.y && idz < c_Size.z)
    {
        output[id] = input[id] * beta;
    }
}