#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subtract_kernel(float* __restrict__ in1, float* __restrict__ in2, float* __restrict__ out)
{
    // Calculate global thread ID
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;
    
    // Ensure thread is within bounds
    if (i < c_Size.x && j < c_Size.y && k < c_Size.z) {
        long int id = (k * c_Size.y + j) * c_Size.x + i;
        out[id] = in1[id] - in2[id];
    }
}