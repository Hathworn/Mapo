#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gradient_and_subtract_kernel(float * in, float * grad_x, float * grad_y, float * grad_z)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i >= c_Size.x || j >= c_Size.y || k >= c_Size.z)
        return;

    long int id = (k * c_Size.y + j) * c_Size.x + i;

    // Use shared memory for coalesced global memory access
    extern __shared__ float s_data[];
    s_data[threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x] = in[id];
    __syncthreads();

    float val = s_data[threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x];
    if (i != (c_Size.x - 1)) {
        float neighbor_x = s_data[threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x + 1];
        grad_x[id] -= ((neighbor_x - val) / c_Spacing.x);
    }
    if (j != (c_Size.y - 1)) {
        float neighbor_y = s_data[threadIdx.z * blockDim.y * blockDim.x + (threadIdx.y + 1) * blockDim.x + threadIdx.x];
        grad_y[id] -= ((neighbor_y - val) / c_Spacing.y);
    }
    if (k != (c_Size.z - 1)) {
        float neighbor_z = s_data[(threadIdx.z + 1) * blockDim.y * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x];
        grad_z[id] -= ((neighbor_z - val) / c_Spacing.z);
    }
}