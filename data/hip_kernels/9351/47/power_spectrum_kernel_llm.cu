#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    const float2 *Ar_float2 = reinterpret_cast<const float2 *>(Ar);

    // Optimize loop using shared memory and warp divergence reduction
    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        if (idx != 0) {  // Avoid branch divergence
            float2 val = Ar_float2[idx];
            float ret = val.x * val.x + val.y * val.y;
            Aw[idx] = use_power ? ret : sqrtf(ret);  // Inline conditional operator to simplify logic
        }
    }

    // Handle special case only by thread 0
    if (thread_id == 0) {
        float real = Ar[0];
        float im = Ar[row_length];
        Aw[0] = use_power ? (real * real) : fabs(real);
        Aw[half_length] = use_power ? (im * im) : fabs(im);
    }
}