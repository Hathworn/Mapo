#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Calculate global indices for the current thread
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Process by rows and columns using grid stride loops
    for (; i < num_rows; i += blockDim.y * gridDim.y) {
        for (int col = j; col < num_cols; col += blockDim.x * gridDim.x) {
            float f = feats[i * ldf + col];
            feats_sq[i * lds + col] = f * f;
        }
    }
}