#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;
    
    float *feats = features + frame * ldf;
    
    // Apply lifter coefficients
    if (cepstral_lifter != 0.0f) {
        int index = thread_id;
        int stride = blockDim.x; // Use blockDim.x instead of CU1DBLOCK for portability
        while (index < num_cols) {
            float lift = lifter_coeffs[index];
            float f = feats[index];
            feats[index] = f * lift;
            index += stride;
        }
    }

    // Thread 0 for each frame will apply energy
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = log(energy_floor);

        // Apply energy flooring condition
        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        feats[0] = energy;
    }
}