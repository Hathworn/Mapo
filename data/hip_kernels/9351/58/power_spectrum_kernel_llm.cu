#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    const float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;
    // Use local variable to reduce repetitive calculation
    int grid_stride = gridDim.x * blockDim.x;

    // Improved loop to handle more data per thread
    for (int idx = thread_id; idx < half_length; idx += grid_stride) {
        // Ignore special case
        if (idx == 0) continue;

        float2 val = reinterpret_cast<const float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        if (use_power) {
            Aw[idx] = ret;
        } else {
            Aw[idx] = sqrtf(ret);
        }
    }

    // Handle special case using block first thread
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabs(real);
            Aw[half_length] = fabs(im);
        }
    }
}