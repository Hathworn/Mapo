#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"



//no performance difference if using float Mono input instead of float4 RGBA
//texture<float, hipTextureType2D, hipReadModeElementType> inTex;
//g_odata[offset] = tex2D(inTex, xc, yc);

texture<float4, hipTextureType2D, hipReadModeElementType> inTex;
surface<void, cudaSurfaceType2D> outputSurface;

__global__ void dataToArray(uint* indices, int imgw, int imgh)
{

int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

int offset = x + y * imgw;

if (x < imgw && y < imgh) {

float res = indices[offset];
//surf2Dwrite(make_float4(res, 0, 0, 1), outputSurface, (int)sizeof(float4)*x, y);
surf2Dwrite(res, outputSurface, (int)sizeof(float)*x, y);
}

}