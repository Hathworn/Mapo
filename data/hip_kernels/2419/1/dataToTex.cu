#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"



//no performance difference if using float Mono input instead of float4 RGBA
//texture<float, hipTextureType2D, hipReadModeElementType> inTex;
//g_odata[offset] = tex2D(inTex, xc, yc);

texture<float4, hipTextureType2D, hipReadModeElementType> inTex;
surface<void, cudaSurfaceType2D> outputSurface;

__global__ void dataToTex(uint* indices, float4 *g_odata, int imgw, int imgh)
{

int x = threadIdx.x + blockIdx.x * blockDim.x;
int y = threadIdx.y + blockIdx.y * blockDim.y;

int offset = x + y * imgw;

if (x < imgw && y < imgh) {

float res = indices[offset];
g_odata[offset] = make_float4(res, 0, 0, 1);
}

}