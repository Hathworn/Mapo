#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define num_thread 256
#define num_block 256

__global__ void blending_pairs(float *a, float *b, float *c, float *d, float *wei, int width, int height, int w, float A, float error_lm, float error_mm, int class_num) {
    // Calculate unique index for each thread
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int Idx = num_thread * bid + tid;

    // Declare variables
    float r_LM, r_MM, r_center_LM, r_center_MM;
    int row, column;
    float sum1, sum2;
    float st = 0.0;
    int judge;
    float dis;
    float weih, result;
    int kk = 0;
    int rmin, rmax, smin, smax;

    // Iterate over the pixels assigned to each thread
    for (int kkk = Idx; kkk < width * height; kkk += num_thread * num_block) {
        result = 0;
        judge = 0;
        wei[kkk] = 0;
        kk = 0;
        sum1 = 0, sum2 = 0;
        row = kkk / width;
        column = kkk % width;
        
        // Calculate central errors
        r_center_LM = d[kkk] - b[kkk] + error_lm;
        r_center_MM = d[kkk] - c[kkk] + 1.412 * error_mm;
        
        // Determine region boundaries
        rmin = max(0, column - w / 2);
        rmax = min(width - 1, column + w / 2);
        smin = max(0, row - w / 2);
        smax = min(height - 1, row + w / 2);

        // Calculate standard deviation within the region
        for (int i = smin; i <= smax; i++) {
            for (int j = rmin; j <= rmax; j++) {
                sum1 += b[i * width + j] * b[i * width + j];
                sum2 += b[i * width + j];
            }
        }
        st = sqrt(sum1 / (w * w) - (sum2 / (w * w)) * (sum2 / (w * w))) / class_num;
        
        // Iterate over region to blend pixels
        for (int i = smin; i <= smax; i++) {
            for (int j = rmin; j <= rmax; j++) {
                if (fabs(b[kkk] - b[i * width + j]) < st) {
                    r_LM = d[i * width + j] - b[i * width + j];
                    r_MM = d[i * width + j] - c[i * width + j];
                    
                    if ((r_center_LM > 0 && r_LM < r_center_LM) || (r_center_LM < 0 && r_LM > r_center_LM)) {
                        if ((r_center_MM > 0 && r_MM < r_center_MM) || (r_center_MM < 0 && r_MM > r_center_MM)) {
                            r_LM = fabs(r_LM) + 0.0001;
                            r_MM = fabs(r_MM) + 0.0001;
                            
                            if (kkk == i * width + j) judge = 1;
                            dis = sqrt(float((row - i) * (row - i) + (column - j) * (column - j))) / A + 1.0;
                            weih = 1.0 / (dis * r_LM * r_MM);
                            wei[kkk] += weih;
                            result += weih * (c[i * width + j] + b[i * width + j] - d[i * width + j]);
                            kk++;
                        }
                    }
                }
            }
        }
        
        // Assign pixel value based on blending results
        if (kk == 0) {
            a[kkk] = fabs(b[kkk] + c[kkk] - d[kkk]) * 1000;
            wei[kkk] = 1000;
        } else {
            if (judge == 0) {
                dis = 1.0;
                r_LM = fabs(d[kkk] - b[kkk]) + 0.0001;
                r_MM = fabs(d[kkk] - c[kkk]) + 0.0001;
                weih = 1.0 / (dis * r_LM * r_MM);
                result += weih * (b[kkk] + c[kkk] - d[kkk]);
                wei[kkk] += weih;
            }
            a[kkk] = result;
        }
    }
}