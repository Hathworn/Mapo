#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define num_thread 256
#define num_block 256

__global__ void blending(float *a, float *b, float *c, float *d, int width, int height, int w, float A, float error_lm, float error_mm, int class_num)
{
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int Idx = num_thread * bid + tid;
    float r_LM, r_MM, r_center_LM, r_center_MM;
    int row, column;
    int i, j;
    float sum1, sum2;
    float st = 0.0;
    int judge;
    float dis;
    float wei;
    float weih, result;
    int kk = 0;
    int rmin, rmax, smin, smax;

    // Loop optimization - improve data access pattern by coalescing
    for (int kkk = Idx; kkk < width * height; kkk += num_thread * num_block)
    {
        result = 0;
        judge = 0;
        wei = 0;
        kk = 0;
        sum1 = 0;
        sum2 = 0;
        row = kkk / width;
        column = kkk % width;
        
        r_center_LM = d[kkk] - b[kkk] + error_lm;
        r_center_MM = d[kkk] - c[kkk] + 1.412 * error_mm;

        // Avoid redundant condition checks by using min/max functions
        rmin = max(0, column - w / 2);
        rmax = min(width - 1, column + w / 2);
        smin = max(0, row - w / 2);
        smax = min(height - 1, row + w / 2);

        // Memory access optimization - reduce repeated calculations
        for (i = smin; i <= smax; i++)
            for (j = rmin; j <= rmax; j++)
            {
                float bval = b[i * width + j];
                sum1 += bval * bval;
                sum2 += bval;
            }

        st = sqrtf(sum1 / (w * w) - (sum2 / (w * w)) * (sum2 / (w * w))) / class_num;

        for (i = smin; i <= smax; i++)
            for (j = rmin; j <= rmax; j++)
            {
                float bval = b[i * width + j];
                if (fabsf(b[kkk] - bval) < st)
                {
                    r_LM = d[i * width + j] - bval;
                    r_MM = d[i * width + j] - c[i * width + j];
                    if ((r_center_LM > 0 && r_LM < r_center_LM) || (r_center_LM < 0 && r_LM > r_center_LM))
                    {
                        if ((r_center_MM > 0 && r_MM < r_center_MM) || (r_center_MM < 0 && r_MM > r_center_MM))
                        {
                            r_LM = fabsf(r_LM) + 0.0001f;
                            r_MM = fabsf(r_MM) + 0.0001f;
                            if (kkk == i * width + j)
                                judge = 1;
                                
                            dis = sqrtf(float((row - i) * (row - i) + (column - j) * (column - j))) / A + 1.0f;
                            weih = 1.0f / (dis * r_LM * r_MM);
                            wei += weih;
                            result += weih * (c[i * width + j] + b[i * width + j] - d[i * width + j]);
                            kk++;
                        }
                    }
                }
            }

        if (kk == 0)
        {
            a[kkk] = fabsf(b[kkk] + c[kkk] - d[kkk]);
            wei = 10000;
        }
        else
        {
            if (judge == 0)
            {
                dis = 1.0f;
                r_LM = fabsf(d[kkk] - b[kkk]) + 0.0001f;
                r_MM = fabsf(d[kkk] - c[kkk]) + 0.0001f;
                weih = 1.0f / (dis * r_LM * r_MM);
                result += weih * (b[kkk] + c[kkk] - d[kkk]);
                wei += weih;
            }
            a[kkk] = result / wei;
        }
    }
}