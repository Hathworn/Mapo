#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use __launch_bounds__ to optimize resource allocation
__launch_bounds__(NUM_RND_THREADS_PER_BLOCK) 
__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    const uint tidx = NUM_RND_THREADS_PER_BLOCK * blockIdx.x + threadIdx.x;
    // Improved initialization using unique seed per block and thread
    hiprand_init(seed + blockIdx.x, tidx, 0, &state[tidx]);
}