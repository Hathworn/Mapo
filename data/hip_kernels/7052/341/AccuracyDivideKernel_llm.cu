#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to divide accuracy by N
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global index
    if (idx == 0) { // Ensure only one thread performs the division
        *accuracy /= N;
    }
}