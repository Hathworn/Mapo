#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    // Compute element index
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Loop through all elements sequentially to compute partial dot products
    if (eidx < numElements) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            sum += a[eidx] * b[eidx];
        }
    }

    // Store computed sum in shared memory
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Perform reduction in shared memory using threads from the block
    if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    __syncthreads();
    if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    __syncthreads();

    // Unroll last warp and use volatile to prevent optimization issues
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    // Store result in target array
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}