#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate thread's global index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    // Loop through all elements for each thread
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Efficient memory access using single calculation for source index
        tgt[i] = __ldg(&src[srcY * srcWidth + srcX]);
    }
}