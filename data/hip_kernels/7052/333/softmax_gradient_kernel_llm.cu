#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    // Use extern shared memory for dynamic allocations
    extern __shared__ float reduction_buffer[];

    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    const int block_size = blockDim.x;
    float tmp = 0.0f;

    // Efficient parallel reduction to compute inner products using shared memory
    for (int i = idx; i < dim; i += block_size) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = block_size / 2; stride > 0; stride /= 2) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    tmp = reduction_buffer[0];

    // Compute gradient
    for (int i = idx; i < dim; i += block_size) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}