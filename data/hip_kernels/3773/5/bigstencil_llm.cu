#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bigstencil(int* in, int* out) {
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we don't access out of bounds memory
    if (i < gridDim.x * blockDim.x) {
        out[i] = in[i] + 2;
    }
}