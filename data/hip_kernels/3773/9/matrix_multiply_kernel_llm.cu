#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    // Cache result variable in registers
    double out = 0.;
    long i = threadIdx.x + blockIdx.x * blockDim.x;

    // Calculate only if within bounds
    if (i < dim_mn) {
        // Use shared memory to reduce global memory access times
        __shared__ double shared_vector_in[1024];

        // Load data into shared memory
        if (threadIdx.x < dim_mn) {
            shared_vector_in[threadIdx.x] = vector_in[threadIdx.x];
        }
        __syncthreads();

        // Loop unrolling for better performance
        for (long j = 0; j < dim_mn; j += 4) {
            out += matrix[i * dim_mn + j] * shared_vector_in[j]
                 + matrix[i * dim_mn + (j + 1)] * shared_vector_in[j + 1]
                 + matrix[i * dim_mn + (j + 2)] * shared_vector_in[j + 2]
                 + matrix[i * dim_mn + (j + 3)] * shared_vector_in[j + 3];
        }
        
        // Write result back to global memory
        vector_out[i] = out;
    }
}