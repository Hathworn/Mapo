#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];

    // Load elements into shared memory
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    tmp[threadIdx.x] = input[index];
    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            tmp[threadIdx.x] += tmp[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (threadIdx.x == 0) {
        output[blockIdx.x] = tmp[0];
    }
}