#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_add(float* a, float* b, int n)
{
    // Utilize a stride loop to let each thread process multiple elements
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (; i < n; i += stride) {
        a[i] += b[i];
    }
}