#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_rndrd_kernel(int *buffer, size_t reps, size_t steps, size_t elements)
{
    // Optimize: Restrict loop iterations and unroll for better throughput
    int errors = 0;
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < elements) {  // Prevents out-of-bound access
        for (size_t j = 0; j < reps; ++j) {
            size_t p = tid;
            size_t a = 548191;
            size_t v = 24819 + (p >> 5);

            #pragma unroll 8  // Unroll loop to increase ILP and hide latency
            for (size_t i = 0; i < steps; i += 4) {
                size_t p0 = p;
                p = (p + (v << 5)) % elements;
                v = (v + a) % elements;
                size_t p1 = p;
                p = (p + (v << 5)) % elements;
                v = (v + a) % elements;
                size_t p2 = p;
                p = (p + (v << 5)) % elements;
                v = (v + a) % elements;
                size_t p3 = p;
                p = (p + (v << 5)) % elements;
                v = (v + a) % elements;

                int v0 = buffer[p0];
                int v1 = buffer[p1];
                int v2 = buffer[p2];
                int v3 = buffer[p3];

                if (v0 != p1) errors++;
                if (v1 != p2) errors++;
                if (v2 != p3) errors++;
                if (v3 != p) errors++;
            }
        }
        if ((errors > 0) && (reps > elements))
            atomicAdd(&buffer[0], errors);  // Use atomic to avoid race conditions
    }
}