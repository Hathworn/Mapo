#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_rndwr_kernel(int *buffer, size_t reps, size_t steps, size_t elements)
{
    // Use shared memory to cache elements accessed within a warp
    extern __shared__ int sharedBuffer[];

    // Calculate global thread ID
    size_t tId = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit for threads outside valid range
    if(tId >= elements) return;

    // Initialization of constants for stepping, accessed once per warp
    size_t a = 548191;
    size_t v = 24819 + (threadIdx.x >> 5);  // Ensure velocity is different for each warp

    // Loop over repetitions
    for(size_t j = 0; j < reps; j++) {
        // Cache data from global to shared memory for current thread
        size_t p = tId;  // Start naturally aligned
        sharedBuffer[threadIdx.x] = buffer[p];

        // Inner stepping loop
        for(size_t i = 0; i < steps; i++) {
            size_t prev = p;
            p = (p + (v << 5)) % elements;  // Maintain warp convergence
            v = (v + a) % elements;        // Velocity update
            buffer[prev] = p;              // Store result back to global memory
        }
    }
}