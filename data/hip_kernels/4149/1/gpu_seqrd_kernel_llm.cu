#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_seqrd_kernel(int *buffer, size_t reps, size_t elements)
{
    int errors = 0;
    size_t ofs = blockIdx.x * blockDim.x + threadIdx.x;
    size_t step = blockDim.x * gridDim.x;

    for(size_t j = 0; j < reps; j++) {
        for (size_t idx = ofs; idx < elements; idx += step * 4) { // Process four elements per loop iteration
            // Manually unroll loop to get multiple loads in flight per thread, using vectorized access
            int val1 = buffer[idx];
            int val2 = (idx + step < elements) ? buffer[idx + step] : 0;
            int val3 = (idx + step * 2 < elements) ? buffer[idx + step * 2] : 0;
            int val4 = (idx + step * 3 < elements) ? buffer[idx + step * 3] : 0;
            // Check result of all the reads
            if(val1 != 0) errors++;
            if(val2 != 0) errors++;
            if(val3 != 0) errors++;
            if(val4 != 0) errors++;
        }
    }
    if(errors > 0)
        atomicAdd(&buffer[0], errors); // Use atomicAdd for potential multi-block errors update
}