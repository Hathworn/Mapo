#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_latency_setup_kernel(int *buffer, size_t delta, size_t elements)
{
    // Calculate global thread index
    size_t ofs = blockIdx.x * blockDim.x + threadIdx.x;

    // Use modulo operation to get target index directly
    while(ofs < elements) {
        buffer[ofs] = (ofs + delta) % elements;
        ofs += blockDim.x * gridDim.x;
    }
}