#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_seqwr_kernel(int *buffer, size_t reps, size_t elements)
{
    size_t step = blockDim.x * gridDim.x; // Precompute step outside the loop for efficiency
    size_t ofs = blockIdx.x * blockDim.x + threadIdx.x; // Calculate initial offset

    for(size_t j = 0; j < reps; j++) {
        for(size_t index = ofs; index < elements; index += step) { // Unroll inner loop for better performance
            buffer[index] = 0;
        }
    }
}