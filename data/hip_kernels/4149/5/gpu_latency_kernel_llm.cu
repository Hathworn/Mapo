#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_latency_kernel(int *buffer, size_t reps, size_t steps, size_t elements)
{
    int errors = 0;

    int p_init = threadIdx.x; // Initialize unique p for each thread
    for (size_t j = 0; j < reps; j++) {
        int p = p_init; // Use the initial p value

        for (size_t i = 0; i < steps; i++) {
            int next = __ldg(&buffer[p]); // Use __ldg for read-only data optimization

            if ((next >= 0) && (next < elements)) {
                p = next;
            } else {
                atomicAdd(&errors, 1); // Use atomic operation to handle errors concurrently
            }
        }
    }

    if ((errors > 0) && (reps > elements))
        atomicExch(&buffer[0], errors); // Use atomicExch to update buffer safely
}