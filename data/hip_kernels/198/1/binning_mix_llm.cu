#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Constants */

#define threads 256

/* Kernels */

__global__ void binning_mix(float *xd_real, float *yd_real, float *zd_real, float *xd_sim, float *yd_sim, float *zd_sim, float *ZY, int lines_number_1, int lines_number_2, int points_per_degree, int number_of_degrees)
{
    __shared__ float temp[threads];
    int index = threadIdx.x;
    temp[index] = 0;
    
    float x, y, z;
    float xx, yy, zz;
    float angle;
    
    int dim_idx;
    int stride = blockDim.x * gridDim.x; // Avoid recomputing stride each time

    for (int i = 0; i < lines_number_1; i++)
    {
        x = xd_real[i];
        y = yd_real[i];
        z = zd_real[i];

        for (dim_idx = blockIdx.x * blockDim.x + threadIdx.x; dim_idx < lines_number_2; dim_idx += stride)
        {
            xx = xd_sim[dim_idx];
            yy = yd_sim[dim_idx];
            zz = zd_sim[dim_idx];

            angle = fminf(x * xx + y * yy + z * zz, 1.0); // Consolidate fminf and dot product
            angle = acosf(angle) * 180.0 / M_PI;

            if (angle < number_of_degrees)
            {
                atomicAdd(&temp[int(angle * points_per_degree)], 1.0);
            }
        }
    }

    atomicAdd(&ZY[index], temp[index]); // Avoid unnecessary synchronization
}