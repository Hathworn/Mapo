#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Constants */
#define threads 256 /* It's the number of threads we are going to use per block on the GPU */

__global__ void binning(float *xd, float *yd, float *zd, float *ZZ, int number_lines, int points_per_degree, int number_of_degrees)
{
    // Use shared memory for temporary results
    __shared__ float temp[threads];
    
    int index = threadIdx.x;
    temp[index] = 0.0f;

    // Calculate global index once
    int dim_idx = blockIdx.x * blockDim.x + index;

    // Loop over the data lines
    for (int i=0; i<number_lines; i++)
    {
        float x = xd[i];
        float y = yd[i];
        float z = zd[i];

        // Ensure dim_idx is within bounds
        while(dim_idx < number_lines)
        {
            float xx = xd[dim_idx];
            float yy = yd[dim_idx];
            float zz = zd[dim_idx];

            // Compute angle using dot product
            float angle = x * xx + y * yy + z * zz;

            /* Correct precision errors */
            angle = fminf(angle, 1.0f);
            angle = acosf(angle) * 180.0f / M_PI;

            // Count pairs within the specified angle range
            if (angle < number_of_degrees)
            {
                atomicAdd(&temp[int(angle * points_per_degree)], 1.0f);
            }

            // Move to the next block of data
            dim_idx += blockDim.x * gridDim.x;
        }
        __syncthreads();
    }

    // Write results from shared to global memory
    atomicAdd(&ZZ[index], temp[index]);
}