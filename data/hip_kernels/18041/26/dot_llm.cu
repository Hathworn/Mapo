#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void dot( int size, float *a, float *b, float *c ) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    // Initialize shared memory to zero
    cache[cacheIndex] = 0.0f;

    // Loop over elements and accumulate product directly in shared memory
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        cache[cacheIndex] += a[i] * b[i];
    }

    // Synchronize threads in this block
    __syncthreads();

    // Reduction using shared memory
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    // Store result for this block
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}