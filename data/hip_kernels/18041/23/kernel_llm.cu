#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel( unsigned char *ptr, int ticks ) {
    // Optimized calculation of pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // Precompute constants for efficiency
    float dim_half = DIM/2;
    float fx = x - dim_half;
    float fy = y - dim_half;
    float d = sqrtf(fx * fx + fy * fy);

    // Use fast intrinsic functions and simplified expressions
    float cos_term = cosf(d * 0.1f - ticks * 0.142857f);
    float denominator = d * 0.1f + 1.0f;
    unsigned char grey = (unsigned char)(128.0f + 127.0f * cos_term / denominator);

    // Efficient memory writes using shared memory or registers
    ptr[offset * 4 + 0] = grey;
    ptr[offset * 4 + 1] = grey;
    ptr[offset * 4 + 2] = grey;
    ptr[offset * 4 + 3] = 255;
}