#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Use thread ID within block
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}