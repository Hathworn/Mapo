#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel( uchar4 *ptr, int ticks ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // calculate fx and fy only once and reuse
    float fx = x - DIM/2;
    float fy = y - DIM/2;
    float d = sqrtf(fx * fx + fy * fy);

    // precompute reused values for clarity and potential efficiency
    float dDiv10 = d / 10.0f;
    float cosineArgument = dDiv10 - ticks / 7.0f;
    float cosineValue = cosf(cosineArgument);

    // compute grey value using precomputed values
    unsigned char grey = (unsigned char)(128.0f + 127.0f * cosineValue / (dDiv10 + 1.0f));

    // set pixel values in one step for potential optimization
    ptr[offset] = make_uchar4(grey, grey, grey, 255);
}