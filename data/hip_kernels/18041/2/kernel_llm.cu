#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Optimize by using registers and removing modulo operations
    if (idx < N) {
        int next_idx1 = (idx + 1);
        int next_idx2 = (idx + 2);
        
        // Load data into registers to reduce global memory access
        float a0 = a[idx];
        float a1 = a[next_idx1 < N ? next_idx1 : next_idx1 - 256];
        float a2 = a[next_idx2 < N ? next_idx2 : next_idx2 - 256];
        
        float b0 = b[idx];
        float b1 = b[next_idx1 < N ? next_idx1 : next_idx1 - 256];
        float b2 = b[next_idx2 < N ? next_idx2 : next_idx2 - 256];
        
        // Calculate averages
        float as = (a0 + a1 + a2) / 3.0f;
        float bs = (b0 + b1 + b2) / 3.0f;
        
        // Write result back to global memory
        c[idx] = (as + bs) / 2;
    }
}