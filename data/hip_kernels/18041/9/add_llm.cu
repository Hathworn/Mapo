#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int addem(int a, int b) {
    return a + b;
}

__global__ void add(int a, int b, int *c) {
    // Use blockIdx and threadIdx to ensure parallel execution
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) { // Ensure only one thread writes to *c
        *c = addem(a, b);
    }
}