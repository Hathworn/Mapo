#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo) {
    // Preload the histogram data into shared memory for faster access
    __shared__ unsigned int local_histo[256]; // Assuming 256 bins for simplicity
    if (threadIdx.x < 256) local_histo[threadIdx.x] = 0; // Initialize shared memory
    
    __syncthreads(); // Ensure all threads have initialized the shared memory

    // Calculate the starting index and the offset to the next block that each thread will be processing
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    while (i < size) {
        atomicAdd(&local_histo[buffer[i]], 1); // Update local histogram
        i += stride;
    }

    __syncthreads(); // Ensure all updates to shared memory are complete

    // Merge results back to global memory
    if (threadIdx.x < 256) atomicAdd(&histo[threadIdx.x], local_histo[threadIdx.x]);
}