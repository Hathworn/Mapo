#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        // Use shared memory to optimize the access
        __shared__ float shared_a[256];
        __shared__ float shared_b[256];

        int localIdx = threadIdx.x;
        shared_a[localIdx] = a[idx];
        shared_b[localIdx] = b[idx];
        __syncthreads();

        int idx1 = (localIdx + 1) % 256;
        int idx2 = (localIdx + 2) % 256;

        float as = (shared_a[localIdx] + shared_a[idx1] + shared_a[idx2]) / 3.0f;
        float bs = (shared_b[localIdx] + shared_b[idx1] + shared_b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}