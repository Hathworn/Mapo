#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // Use global thread ID
    if (tid < n)  // Ensure thread is within bounds
        c[tid] = a[tid] + b[tid];
}