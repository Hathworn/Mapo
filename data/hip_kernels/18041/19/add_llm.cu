#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add( int *a, int *b, int *c ) {
    // Calculate global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x; 
    int stride = blockDim.x * gridDim.x;
    
    // Use loop unrolling to optimize performance
    while (tid + 4 < N) {
        c[tid] = a[tid] + b[tid];
        c[tid + 1] = a[tid + 1] + b[tid + 1];
        c[tid + 2] = a[tid + 2] + b[tid + 2];
        c[tid + 3] = a[tid + 3] + b[tid + 3];
        tid += 4 * stride;
    }
    
    // Process remaining elements if N is not divisible by 4*stride
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += stride;
    }
}