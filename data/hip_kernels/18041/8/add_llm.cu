#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c) {
    // Calculate global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a conditional check to reduce unnecessary memory writes
    if(idx == 0) {
        *c = a + b;
    }
}