#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot(float *a, float *b, float *c) {
    extern __shared__ float cache[]; // Use dynamic shared memory

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    float temp = 0;

    // Each thread computes partial dot product
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        temp += a[i] * b[i];
    }

    // Store result in cache
    cache[cacheIndex] = temp;
    __syncthreads();

    // Parallel reduction in shared memory
    for (int i = blockDim.x / 2; i > 0; i /= 2) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
    }

    // Write result for block to global memory
    if (cacheIndex == 0) {
        c[blockIdx.x] = cache[0];
    }
}