#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histo_kernel(unsigned char *buffer, long size, unsigned int *histo) {
    // Declare and clear shared memory
    __shared__ unsigned int temp[256];
    temp[threadIdx.x] = 0;
    __syncthreads();

    // Calculate starting index and stride
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Use loop unrolling for efficiency
    while (i < size) {
        atomicAdd(&temp[buffer[i]], 1);
        i += stride;
    }
    __syncthreads();

    // Use one atomic add per thread to update global histogram
    atomicAdd(&(histo[threadIdx.x]), temp[threadIdx.x]);
}