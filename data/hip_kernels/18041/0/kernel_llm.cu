#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)

__global__ void kernel(int *a, int *b, int *c) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) & 255; // Use bitwise AND instead of modulo for power of 2
        int idx2 = (idx + 2) & 255; // Use bitwise AND instead of modulo for power of 2

        // Load values from global memory to registers
        float a0 = a[idx];
        float a1 = a[idx1];
        float a2 = a[idx2];
        float b0 = b[idx];
        float b1 = b[idx1];
        float b2 = b[idx2];

        // Compute averaged sums
        float as = (a0 + a1 + a2) * 0.333333f; // Use multiplication instead of division
        float bs = (b0 + b1 + b2) * 0.333333f;

        c[idx] = (as + bs) * 0.5f; // Simplify expression
    }
}