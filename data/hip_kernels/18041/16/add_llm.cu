#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID
    int stride = gridDim.x * blockDim.x; // Calculate stride for entire grid
    while (tid < N) {
        c[tid] = a[tid] + b[tid]; // Perform addition
        tid += stride; // Move to the next element for this thread
    }
}