#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuConvert32uC1To32fC1Kernel(const unsigned int *src, size_t src_stride, float* dst, size_t dst_stride, float mul_constant, float add_constant, int width, int height)
{
    // Calculate global thread position
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Use shared memory for strided access
    __shared__ unsigned int shared_src[32][32];
    __shared__ float shared_dst[32][32];

    // Convert linear indexes to 2D block coordinates
    int src_c = y * src_stride + x;
    int dst_c = y * dst_stride + x;

    // Load elements into shared memory to improve coalesced memory access
    if (x < width && y < height) {
        shared_src[threadIdx.y][threadIdx.x] = src[src_c];
    }

    // Synchronize threads to ensure all shared memory operations are complete
    __syncthreads();

    if (x < width && y < height) {
        // Perform the conversion using shared memory
        shared_dst[threadIdx.y][threadIdx.x] = shared_src[threadIdx.y][threadIdx.x] * mul_constant + add_constant;
        dst[dst_c] = shared_dst[threadIdx.y][threadIdx.x];
    }
}