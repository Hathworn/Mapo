#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuConvertRGBToHSVKernel(const float4* src, float4* dst, size_t stride, int width, int height, bool normalize)
{
    // Calculate pixel position
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int c = y * stride + x;

    if (x < width && y < height)
    {
        // Read input color
        float4 in = src[c];
        float R = in.x;
        float G = in.y;
        float B = in.z;

        // Calculate max, min, and delta
        float Ma = fmaxf(R, fmaxf(G, B));
        float mi = fminf(R, fminf(G, B));
        float C = Ma - mi;

        // Initialize Hue, Saturation, Value
        float H = 0.0f;
        float S = 0.0f;
        float V = Ma;

        // Compute Hue
        if (C != 0.0f)
        {
            if (Ma == R)
                H = fmodf((G - B) / C, 6.0f);
            else if (Ma == G)
                H = (B - R) / C + 2.0f;
            else
                H = (R - G) / C + 4.0f;
            
            H *= 60.0f;
            if (H < 0.0f)
                H += 360.0f;
            
            // Compute Saturation
            S = C / V;

            // Normalize if required
            if (normalize)
                H /= 360.0f;
        }

        // Write converted HSV to destination
        dst[c] = make_float4(H, S, V, in.w);
    }
}