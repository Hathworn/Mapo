#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    // Calculate global index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use warp-synchronous programming to reduce branching
    int warp_idx = i / warpSize;
    int lane_idx = i % warpSize;

    // Ensure thread and warp index calculation minimizes divergence
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}