#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuConvertRGBToLABKernel(const float4* src, float4* dst, size_t stride, int width, int height, bool isNormalized)
{
    // Calculate the global index for the current thread
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int c = y * stride + x;

    if (x < width && y < height)
    {
        // Read input pixel
        float4 in = src[c];
        
        // Normalize RGB components if not already normalized
        if (!isNormalized)
        {
            in.x *= 0.003921569f; // Division by 255.0f
            in.y *= 0.003921569f; // Division by 255.0f
            in.z *= 0.003921569f; // Division by 255.0f
        }

        const float R = in.x;
        const float G = in.y;
        const float B = in.z;

        // Convert RGB to XYZ color space
        float4 XYZ;
        XYZ.x = 0.4124564f * R + 0.3575761f * G + 0.1804375f * B;
        XYZ.y = 0.2126729f * R + 0.7151522f * G + 0.0721750f * B;
        XYZ.z = 0.0193339f * R + 0.1191920f * G + 0.9503041f * B;

        // Normalize XYZ using D65 white point
        XYZ.x *= 1.0521267f; // Division by 0.950456f
        XYZ.z *= 0.9183173f; // Division by 1.088754f

        const float T1 = 0.008856f; // 216/24389
        const float T2 = 7.787f;    // 24389/27 = (24389/(27*116))

        // Compute L, a, b components using the cube root condition
        const float fx = (XYZ.x > T1) ? cbrtf(XYZ.x) : (T2 * XYZ.x + 0.137931); // Division by 116
        const float fy = (XYZ.y > T1) ? cbrtf(XYZ.y) : (T2 * XYZ.y + 0.137931); // Division by 116
        const float fz = (XYZ.z > T1) ? cbrtf(XYZ.z) : (T2 * XYZ.z + 0.137931); // Division by 116

        // Write resulting LAB pixel
        dst[c] = make_float4(116.0f * fy - 16.0f, 500.0f * (fx - fy), 200.0f * (fy - fz), in.w);
    }
}