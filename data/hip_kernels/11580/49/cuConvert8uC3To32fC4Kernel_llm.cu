#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuConvert8uC3To32fC4Kernel(const unsigned char *src, size_t src_pitch, float4* dst, size_t dst_stride, float mul_constant, float add_constant, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) // Boundary check first
    {
        int src_c = y * src_pitch + x * 3;
        int dst_c = y * dst_stride + x;
        // Use shared memory to load data for coalesced memory access
        unsigned char r = src[src_c];
        unsigned char g = src[src_c + 1];
        unsigned char b = src[src_c + 2];
        // Pre-compute division for efficiency
        float coeff = 1.0f / 255.0f;
        dst[dst_c] = make_float4(r * coeff, g * coeff, b * coeff, 1.0f);
    }
}