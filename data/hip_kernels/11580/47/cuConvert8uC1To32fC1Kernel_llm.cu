#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuConvert8uC1To32fC1Kernel(const unsigned char *src, size_t src_stride, float* dst, size_t dst_stride, float mul_constant, float add_constant, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Combine index calculation into a single operation to reduce instruction count
    size_t src_c = static_cast<size_t>(y) * src_stride + x;
    size_t dst_c = static_cast<size_t>(y) * dst_stride + x;

    // Check bounds before performing operation to avoid out-of-bounds access
    if (x < width && y < height) {
        dst[dst_c] = static_cast<float>(src[src_c]) * mul_constant + add_constant;
    }
}