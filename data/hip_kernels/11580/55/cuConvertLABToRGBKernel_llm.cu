#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuConvertLABToRGBKernel(const float4* __restrict__ src, float4* __restrict__ dst, size_t stride, int width, int height)
{
    // Calculate global thread coordinates
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int c = y * stride + x;

    // Process only if within image bounds
    if (x < width && y < height)
    {
        // Read source data
        float4 in = src[c];

        float L = in.x;
        float a = in.y;
        float b = in.z;

        // Convert to XYZ
        const float T1 = cbrtf(216 / 24389.0f);
        const float fy = (L + 16) / 116.0f;

        float4 XYZ;
        if (L > 8)
            XYZ.y = fy * fy * fy;
        else
            XYZ.y = L / (24389 / 27.0f);

        float fx = a / 500.0f + fy;
        if (fx > T1)
            XYZ.x = fx * fx * fx;
        else
            XYZ.x = (116 * fx - 16) / (24389 / 27.0f);

        float fz = fy - b / 200.0f;
        if (fz > T1)
            XYZ.z = fz * fz * fz;
        else
            XYZ.z = (116 * fz - 16) / (24389 / 27.0f);

        // Normalize for D65 white point
        XYZ.x *= 0.950456f;
        XYZ.z *= 1.088754f;

        // Linear transformation to RGB
        float4 rgb;
        rgb.x = fmaf(3.2404542f, XYZ.x, fmaf(-1.5371385f, XYZ.y, -0.4985314f * XYZ.z));
        rgb.y = fmaf(-0.9692660f, XYZ.x, fmaf(1.8760108f, XYZ.y, 0.0415560f * XYZ.z));
        rgb.z = fmaf(0.0556434f, XYZ.x, fmaf(-0.2040259f, XYZ.y, 1.0572252f * XYZ.z));
        rgb.w = in.w;

        // Write to destination
        dst[c] = rgb;
    }
}