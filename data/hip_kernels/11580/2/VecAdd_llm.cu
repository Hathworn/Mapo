#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory for performance improvement
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Check bounds before accessing in case of over-indexing
    if(i < N) {
        C[i] = A[i] + B[i];
    }
}