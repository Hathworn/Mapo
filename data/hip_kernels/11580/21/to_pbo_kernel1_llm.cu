#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void to_pbo_kernel1(unsigned char* g_in, int stride_in, uchar4* g_out, int stride_out, int width, int height)
{
    // Calculate global thread indices
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if within image bounds
    if (x < width && y < height)
    {
        // Read and write to global memory
        int index_in = y * stride_in + x;      // Compute input index once
        unsigned char value = g_in[index_in];  // Single read operation
        int index_out = y * stride_out + x;    // Compute output index once
        g_out[index_out] = make_uchar4(value, value, value, 1); // Use computed index
    }
}