#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuConvertHSVToRGBKernel(const float4* src, float4* dst, size_t stride, int width, int height, bool denormalize)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Calculate x coordinate within the grid
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Calculate y coordinate within the grid
    
    int c = y * stride + x; // Calculate linear index of the pixel
    
    if (x < width && y < height)
    {
        // Read HSV values
        float4 in = src[c];
        float H = in.x;
        float S = in.y;
        float V = in.z;

        float4 rgb = make_float4(0.0f, 0.0f, 0.0f, 0.0f);

        // If saturation is zero, it's a gray-scale pixel
        if (S == 0) 
        {
            rgb = make_float4(V, V, V, in.w); // Directly use V for RGB components
            dst[c] = rgb; // Write back and return early
            return;
        }

        // Normalize hue
        H /= 60.0f;
        int i = floor(H); // Determine the color sector
        float f = H - i; // Calculate intermediate value
        float p = V * (1.0f - S); // Intermediate value p
        float q = V * (1.0f - S * f); // Intermediate value q
        float t = V * (1.0f - S * (1.0f - f)); // Intermediate value t

        // Conditional assignments based on color sector
        if (i == 0)
            rgb = make_float4(V, t, p, in.w);
        else if (i == 1)
            rgb = make_float4(q, V, p, in.w);
        else if (i == 2)
            rgb = make_float4(p, V, t, in.w);
        else if (i == 3)
            rgb = make_float4(p, q, V, in.w);
        else if (i == 4)
            rgb = make_float4(t, p, V, in.w);
        else if (i == 5)
            rgb = make_float4(V, p, q, in.w);

        // Write Back
        rgb.w = in.w; // Preserve alpha channel
        dst[c] = rgb; // Store the converted RGB values
    }
}