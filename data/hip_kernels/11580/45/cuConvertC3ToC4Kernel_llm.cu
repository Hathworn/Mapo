#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuConvertC3ToC4Kernel(const float3* src, size_t src_stride, float4* dst, size_t dst_stride, int width, int height)
{
    // Calculate the linear index instead of separate x and y
    int idx = blockIdx.y * blockDim.y * src_stride + blockIdx.x * blockDim.x + threadIdx.y * src_stride + threadIdx.x;

    // Ensure index is within bounds
    if (threadIdx.x + blockIdx.x * blockDim.x < width && threadIdx.y + blockIdx.y * blockDim.y < height)
    {
        float3 val = src[idx];
        dst[idx] = make_float4(val.x, val.y, val.z, 1.0f);
    }
}