#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuConvertC4ToC3Kernel(const float4* __restrict__ src, size_t src_stride, float3* __restrict__ dst, size_t dst_stride, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Pre-calculate indices to reduce repetitive calculations
    if (x < width && y < height)
    {
        int src_c = y * src_stride + x;
        int dst_c = y * dst_stride + x;

        // Load float4 once to improve memory coalescing and reduce read transactions
        float4 val = src[src_c];
        dst[dst_c] = make_float3(val.x, val.y, val.z);
    }
}