#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuFilterGaussZKernel_32f_C1(float* dst, float* src, const int y, const int width, const int depth, const size_t stride, const size_t slice_stride, float sigma, int kernel_size)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int z = blockIdx.y * blockDim.y + threadIdx.y;

    if(x < width && z < depth) // Removed unnecessary checks
    {
        float sum = 0.0f;
        int half_kernel_elements = (kernel_size - 1) / 2;

        // Precompute constants for Gaussian
        float g0 = 1.0f / (sqrtf(2.0f * 3.141592653589793f) * sigma);
        float g1 = exp(-0.5f / (sigma * sigma));
        float g2 = g1 * g1;

        sum = g0 * src[z * slice_stride + y * stride + x];
        float sum_coeff = g0;

        float g = g0; // Use 'g' to track the current weight factor
        for (int i = 1; i <= half_kernel_elements; i++)
        {
            g *= g1;
            int cur_z_pos = min(depth - 1, z + i);
            int cur_z_neg = max(0, z - i);
            sum += g * (src[cur_z_pos * slice_stride + y * stride + x] + src[cur_z_neg * slice_stride + y * stride + x]);
            sum_coeff += 2.0f * g;
        }
        dst[z * slice_stride + y * stride + x] = sum / sum_coeff;
    }
}