#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAdd(float* __restrict__ A, float* __restrict__ B, float* __restrict__ C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < N)
    {
        // Load values before use
        float a = A[i];
        float b = B[i];
        C[i] = a + b;
    }
}