#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/* Optimize kernel by reducing branching and improving memory access */

/*numCirs: num of total circles
*/
__global__ void kernelCompact(float* devSrc, float* devDst, unsigned int* devPredicate, unsigned int* devPos, int numCirs, int offset)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x; //index of the circles

    // Check bounds once to avoid unnecessary checks
    if (idx < numCirs && devPredicate[idx] == 1)
    {
        unsigned int pos = devPos[idx] * offset;
        idx *= offset;

        // Optimize memory access pattern by unrolling the loop
        for (int i = 0; i < offset; ++i)
        {
            devDst[pos + i] = devSrc[idx + i];
        }
    }
}