#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelFormBinStart(int* devOutputBinStart, unsigned int* devInputBinCirPairBin, unsigned int bcPairLen)
{
    // Declare shared memory with a dynamic size to fit the block size
    extern __shared__ int cache[];

    int bcPairIdx = blockDim.x * blockIdx.x + threadIdx.x;

    if (bcPairIdx >= bcPairLen)
    {
        return;
    }

    // Load current bin pair values into shared memory
    cache[threadIdx.x + 1] = devInputBinCirPairBin[bcPairIdx];

    // Handle the first thread storing the last bin of the previous block
    if (threadIdx.x == 0)
    {
        if (bcPairIdx != 0)
        {
            cache[0] = devInputBinCirPairBin[bcPairIdx - 1];
        }
        else
        {
            cache[0] = -1;
        }
    }

    __syncthreads();

    // Check for transition and update output based on it
    if (threadIdx.x == 0 || cache[threadIdx.x + 1] != cache[threadIdx.x])
    {
        devOutputBinStart[cache[threadIdx.x + 1]] = bcPairIdx;
    }
}