#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addInc(unsigned int* deviceInput, unsigned int* deviceOutput, int eleCnt, unsigned int* deviceInc)
{
    // Load increment value into shared memory for each block
    __shared__ unsigned int inc;
    if (threadIdx.x == 0) {
        inc = deviceInc[blockIdx.x];
    }
    __syncthreads();

    int cntInB = blockDim.x * 2;
    int idxInG = blockIdx.x * cntInB + threadIdx.x;

    // Use shared memory increment for both indices
    if (idxInG < eleCnt) {
        deviceOutput[idxInG] = deviceInput[idxInG] + inc;
    }

    if (idxInG + blockDim.x < eleCnt) {
        deviceOutput[idxInG + blockDim.x] = deviceInput[idxInG + blockDim.x] + inc;
    }
}