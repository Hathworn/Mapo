#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void preScan(unsigned int* deviceInput, unsigned int* deviceOutput, int cnt, unsigned int* deviceSum)
{
    extern __shared__ unsigned int temp[];
    int cntInB = blockDim.x * 2;
    int idxInG = cntInB * blockIdx.x + threadIdx.x;
    int idxInB = threadIdx.x;

    // Initialize shared memory
    temp[2 * idxInB] = (idxInG < cnt) ? deviceInput[idxInG] : 0;
    temp[2 * idxInB + 1] = (idxInG + blockDim.x < cnt) ? deviceInput[idxInG + blockDim.x] : 0;

    int offset = 1;
    for (int d = cntInB >> 1; d > 0; d >>= 1)
    {
        __syncthreads();
        if (threadIdx.x < d)
        {
            int ai = offset * (2 * threadIdx.x + 1) - 1;
            int bi = ai + offset;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    __syncthreads();
    // Move the sum to deviceSums and reset last element in shared memory
    if (threadIdx.x == 0)
    {
        deviceSum[blockIdx.x] = temp[cntInB - 1];
        temp[cntInB - 1] = 0;
    }

    // Downsweep phase
    for (int d = 1; d < cntInB; d *= 2)
    {
        offset >>= 1;
        __syncthreads();
        if (threadIdx.x < d)
        {
            int ai = offset * (2 * threadIdx.x + 1) - 1;
            int bi = ai + offset;
            unsigned int be = temp[bi];
            temp[bi] += temp[ai];
            temp[ai] = be;
        }
    }

    __syncthreads();
    // Write results to device output
    if (idxInG < cnt)
    {
        deviceOutput[idxInG] = temp[2 * idxInB];
    }
    if (idxInG + blockDim.x < cnt)
    {
        deviceOutput[idxInG + blockDim.x] = temp[2 * idxInB + 1];
    }
}