#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Use threadIdx.x for parallelism within a block
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    c[i] = a[i] + b[i];
}