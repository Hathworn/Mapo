#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Optimize by calculating the global thread index for full utilization
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use (if) condition to prevent out-of-bound access for arrays larger than block size
    if (i < N)  // Assuming N is the total number of elements
    {
        c[i] = a[i] + b[i];
    }
}