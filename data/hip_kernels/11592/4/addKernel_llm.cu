#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate global index based on block and thread indices for scalability
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform addition only if index is within bounds (assuming size of arrays is passed externally)
    // This requires block and grid configuration to be compatible with the array sizes
    if (i < N) 
    {
        c[i] = a[i] + b[i];
    }
}