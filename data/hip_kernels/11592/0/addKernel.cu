#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
int i = blockIdx.x;
c[i] = a[i] + b[i];
}