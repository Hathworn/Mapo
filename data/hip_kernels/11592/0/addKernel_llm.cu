#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);

// Optimize by computing index using blockIdx.x and threadIdx.x for scalability
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < size) // Boundary check to prevent out-of-bounds access
    {
        c[i] = a[i] + b[i]; // Perform addition
    }
}