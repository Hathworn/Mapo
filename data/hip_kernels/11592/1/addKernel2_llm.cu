#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);

__global__ void addKernel2(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use both block and thread index for better parallelization
    c[i] = (a[i] + b[i]) * 2;
}