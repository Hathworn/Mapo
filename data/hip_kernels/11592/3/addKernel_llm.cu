#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a)
{
    int i = threadIdx.x;
    extern __shared__ int smem[];

    // Load input into shared memory in parallel
    smem[i] = a[i];
    __syncthreads();

    // Use if-else if-else statements for better readability and maintainability
    if (i == 0)  // Thread 0 calculates the sum of squares
    {
        int sum_squares = 0;
        for (int d = 0; d < 5; d++)
        {
            sum_squares += smem[d] * smem[d];
        }
        c[0] = sum_squares;
    }
    else if (i == 1) // Thread 1 performs summation
    {
        int sum = 0;
        for (int d = 0; d < 5; d++)
        {
            sum += smem[d];
        }
        c[1] = sum;
    }
    else if (i == 2)  // Thread 2 performs product
    {
        int product = 1;
        for (int d = 0; d < 5; d++)
        {
            product *= smem[d];
        }
        c[2] = product;
    }
}