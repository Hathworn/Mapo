#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Ensure indexes are within bounds
    if (i < h && j < w) {
        const int pos = i * s + j;

        // Simplified conditional scale adjustment with ternary operator
        float scale = normalization_factor[pos];
        float invScale = (scale != 0.0f) ? (1.0f / scale) : 1.0f;

        // Apply normalization directly
        image[pos] *= invScale;
    }
}