#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    
    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);
    
    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];
    float weight;

    // Optimize pixel fill checks
    #pragma unroll
    for (int offset = 0; offset < 4; ++offset)
    {
        int x_offset = (offset == 1 || offset == 3) ? -1 : 0;
        int y_offset = (offset >= 2) ? -1 : 0;
        
        int current_tx = tx + x_offset;
        int current_ty = ty + y_offset;

        if (!(current_tx >= w || current_tx < 0 || current_ty >= h || current_ty < 0))
        {
            weight = ((offset % 2 == 0) ? dx : (1.0f - dx)) * ((offset < 2) ? dy : (1.0f - dy));
            _atomicAdd(dst + current_ty * image_stride + current_tx, value * weight);
            _atomicAdd(normalization_factor + current_ty * image_stride + current_tx, weight);
        }
    }
}