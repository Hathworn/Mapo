#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        short2 loc = loc_[ptidx];

        // Apply scale directly without temporary variables for efficiency
        x[ptidx] = __int2float_rn(loc.x) * scale;
        y[ptidx] = __int2float_rn(loc.y) * scale;
    }
}