#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image) {
    // Calculate a unique global thread ID
    int global_id = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * blockIdx.y);

    // Calculate the total number of threads globally
    int total_threads = blockDim.x * blockDim.y * gridDim.y;

    // Use stride to optimize global memory access
    for (int idx = global_id; idx < w * h; idx += total_threads) {
        image[idx] = value;
    }
}