#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

__global__ void computeDistances(int numInstances, int numAttributes, float* dataset, float* distances)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < numInstances * numInstances)
    {
        int row = tid / numInstances;
        int column = tid % numInstances; // Use modulo to find instance2Index for better readability

        float sum = 0;
        int instance1 = row * numAttributes;
        int instance2 = column * numAttributes;

        /* Loop unrolling to reduce loop overhead and allow better optimization */
        // Assuming that numAttributes is a multiple of 4 for simplification, otherwise adjust the loop appropriately
        for (int atIdx = 1; atIdx < numAttributes; atIdx += 4)
        {
            float diff1 = dataset[instance1 + atIdx] - dataset[instance2 + atIdx];
            float diff2 = dataset[instance1 + atIdx + 1] - dataset[instance2 + atIdx + 1];
            float diff3 = dataset[instance1 + atIdx + 2] - dataset[instance2 + atIdx + 2];
            float diff4 = dataset[instance1 + atIdx + 3] - dataset[instance2 + atIdx + 3];
            sum += diff1 * diff1 + diff2 * diff2 + diff3 * diff3 + diff4 * diff4;
        }

        float dist = sqrtf(sum); // Use sqrtf for better performance with float

        distances[row * numInstances + column] = dist;
        distances[column * numInstances + row] = dist; // Symmetric distance assignment
    }
}