#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void permute_colors_kernel(int num_rows, int *row_colors, int *color_permutation)
{
    int row_id = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize loop to avoid branch divergence and to improve memory coalescing
    while (row_id < num_rows)
    {
        int color = row_colors[row_id];

        // Use of __ldg intrinsic function optimized for newer GPU architectures
        #if __CUDA_ARCH__ >= 350
        color = __ldg(color_permutation + color);
        #else
        color = color_permutation[color];
        #endif

        row_colors[row_id] = color;
        
        row_id += blockDim.x * gridDim.x;
    }
}