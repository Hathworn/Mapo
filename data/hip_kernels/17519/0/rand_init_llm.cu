#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define FALSE 0
#define TRUE  1

// Optimized to ensure tid is within bounds of array
__global__ void rand_init(long *a, long seed, size_t size)
{
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if (tid < size) // Check bounds
    {
        a[tid] = seed + tid; // Use tid directly
    }
}