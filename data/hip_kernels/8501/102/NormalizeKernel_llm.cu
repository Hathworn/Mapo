#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate the global position within the image.
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Check bounds to avoid accessing out of bounds memory.
    if (i < h && j < w)
    {
        const int pos = i * s + j;

        // Read normalization factor and compute inverse scale with ternary operator.
        float scale = normalization_factor[pos];
        float invScale = (scale != 0.0f) ? (1.0f / scale) : 1.0f;

        // Apply normalization to image at the calculated position.
        image[pos] *= invScale;
    }
}