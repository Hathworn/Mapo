#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    // Calculate offsets and values more efficiently
    int offset = i * flow_stride + j;
    float u_ = u[offset];
    float v_ = v[offset];
    float value = src[i * image_stride + j];

    // Calculate target coordinates
    float cx = u_ * time_scale + (float)j + 1.0f;
    float cy = v_ * time_scale + (float)i + 1.0f;
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Validate target coordinates and perform atomic add
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}