#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate the 1D position
    int pos = idy * w + idx;

    // Ensure idx, idy are within bounds
    if (idy < h && idx < w) {
        image[pos] = value;
    }
}