#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void createLookupKernel(const int* inds, int total, int* output)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Optimize condition check to avoid divergency
    if (idx < total) {
        int ind_value = inds[idx]; // Reduce memory access by reusing loaded value
        output[ind_value] = idx;
    }
}