#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Use a grid-stride loop to optimize for larger arrays
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = tid; i < N; i += stride) {
        out[i] = fmaxf(in[i], 0.0f); // Use fmaxf for more clarity
    }
}