#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_copy_rc_cu_z(const double *din, double *zout, const int n) {
    // Calculate linear thread id for entire grid
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride to allow one thread to handle multiple elements
    for (int i = igpt; i < n; i += gridDim.x * blockDim.x) {
        zout[2 * i] = din[i];
        zout[2 * i + 1] = 0.0e0;
    }
}