#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_scatter_cu_z(double *c, const double *pwcc, const double scale, const int ngpts, const int nmaps, const int *ghatmap) {
    // Calculate the global index
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    if (igpt < ngpts) {
        int map_index = ghatmap[igpt];
        // Perform scaled value assignment to c array
        double scaled_real = scale * pwcc[2 * igpt];
        double scaled_imag = scale * pwcc[2 * igpt + 1];
        c[2 * map_index] = scaled_real;
        c[2 * map_index + 1] = scaled_imag;
        
        // Handle the case when nmaps is 2
        if (nmaps == 2) {
            int map_index_nmaps2 = ghatmap[igpt + ngpts];
            c[2 * map_index_nmaps2] = scaled_real;
            c[2 * map_index_nmaps2 + 1] = -scaled_imag;
        }
    }
}