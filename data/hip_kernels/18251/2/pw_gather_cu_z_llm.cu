#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_gather_cu_z(double *pwcc, const double *c, const double scale, const int ngpts, const int *ghatmap) {
    // Calculate global thread index using optimized method
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    if (igpt < ngpts) {
        // Directly calculate the indices to enhance performance
        const int mapIdx = 2 * ghatmap[igpt];
        pwcc[2 * igpt] = scale * c[mapIdx];
        pwcc[2 * igpt + 1] = scale * c[mapIdx + 1];
    }
}