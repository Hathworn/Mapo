#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_copy_cr_cu_z(const double *zin, double *dout, const int n) {
    // Calculate global index more efficiently
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Use an if condition to avoid out of bounds access and unnecessary computation
    if (igpt < n) {
        dout[igpt] = zin[2 * igpt];
    }
}