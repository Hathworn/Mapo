#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VecAdd(float * A, float * B, float * C, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check to ensure within bounds
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}