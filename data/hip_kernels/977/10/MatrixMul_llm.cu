#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMul(int *M, int *N, int *P, int width)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    const int tile_size = 16;

    __shared__ int As[tile_size][tile_size];
    __shared__ int Bs[tile_size][tile_size];

    int aBegin = width * tile_size * by;
    int aEnd   = aBegin + width - 1;
    int aStep  = tile_size;

    int bBegin = tile_size * bx;
    int bStep  = tile_size * width;

    int Csub = 0;

    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
    {
        As[ty][tx] = M[a + width * ty + tx]; // Memory coelescing
        Bs[ty][tx] = N[b + width * ty + tx]; // Memory coelescing
        __syncthreads();

        // Unrolling loop to reduce iteration overhead and avoid bank conflicts
        for (int k = 0; k < tile_size; k += 4) 
        {
            Csub += As[ty][k]   * Bs[tx][k];
            Csub += As[ty][k+1] * Bs[tx][k+1];
            Csub += As[ty][k+2] * Bs[tx][k+2];
            Csub += As[ty][k+3] * Bs[tx][k+3];
        }
        __syncthreads();
    }

    int c = width * tile_size * by + tile_size * bx;
    P[c + width * ty + tx] = Csub;
}