#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *a, int offset)
{
    int i = offset + threadIdx.x + blockIdx.x * blockDim.x;
    float x = static_cast<float>(i);

    // Precompute squares to avoid duplicate calculations
    float s = sinf(x);
    float c = cosf(x);
    float s2_c2 = s * s + c * c;
    
    // Use fmaf for better performance (if applicable)
    a[i] += sqrtf(s2_c2);
}