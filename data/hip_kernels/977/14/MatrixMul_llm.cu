#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMul(int *M, int *N, int *P, int width)
{
    __shared__ int M_shared[blockDim.y][blockDim.x]; // Shared memory for M
    __shared__ int N_shared[blockDim.y][blockDim.x]; // Shared memory for N

    int accu = 0;

    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int i = by * blockDim.y + ty;
    int j = bx * blockDim.x + tx;

    for (int kk = 0; kk < width; kk += blockDim.x) // Iterating in chunks
    {
        // Load M and N into shared memory
        M_shared[ty][tx] = M[i * width + (kk + tx)];
        N_shared[ty][tx] = N[(kk + ty) * width + j];

        __syncthreads(); // Synchronize threads

        for (int k = 0; k < blockDim.x; k++)
        {
            accu += M_shared[ty][k] * N_shared[k][tx];
        }

        __syncthreads(); // Synchronize threads for next iteration
    }

    P[i * width + j] = accu; // Store the result
}