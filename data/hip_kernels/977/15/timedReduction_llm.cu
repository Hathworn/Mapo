#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void timedReduction(const float *input, float *output, clock_t *timer)
{
    extern __shared__ float shared[];

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    if (tid == 0) timer[bid] = clock();

    // Load input to shared memory
    shared[tid] = input[tid];
    shared[tid + blockDim.x] = input[tid + blockDim.x];
    __syncthreads(); // Ensure all loads are completed

    // Perform reduction to find minimum
    for (int d = blockDim.x; d > 0; d /= 2)
    {
        if (tid < d)
        {
            float f0 = shared[tid];
            float f1 = shared[tid + d];

            if (f1 < f0)
            {
                shared[tid] = f1;
            }
        }
        __syncthreads(); // Ensure all computations are done before next iteration
    }

    // Write result
    if (tid == 0) output[bid] = shared[0];

    if (tid == 0) timer[bid + gridDim.x] = clock();
}