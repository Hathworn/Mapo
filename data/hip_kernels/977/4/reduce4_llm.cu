#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

template <unsigned int blockSize>
__device__ void warpReduce(volatile int* sdata, int tid) {
  if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
  if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
  if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
  if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
  if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
  if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

__global__ void reduce4(int *g_idata, int *g_odata) {
  extern __shared__ int sdata[];
  
  unsigned int tid = threadIdx.x;
  unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
  
  // Load data into shared memory using a single read
  sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
  __syncthreads();

  // Reduce using a single loop up to 32 for warpReduce
  for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
    if (tid < s) {
      sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  // Optimized warp reduction
  if (tid < 32) warpReduce<blockDim.x>(sdata, tid);

  // Write the result to global memory
  if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}