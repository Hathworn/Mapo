#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

template <unsigned int blockSize>
__device__ void warpReduce(volatile int* sdata, int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

__global__ void reduce5(int* g_idata, int* g_odata) {
    extern __shared__ int sdata[];
    // Efficient indexing variables
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    
    // Use less divergent memory access
    if (i + blockDim.x < gridDim.x * blockDim.x) {
        sdata[tid] = g_idata[i] + g_idata[i + blockDim.x];
    } else {
        sdata[tid] = g_idata[i]; // Handle incomplete final block
    }
    __syncthreads();

    // Unroll the reduction loop
    if (blockSize >= 512) {
        if (tid < 256) { sdata[tid] += sdata[tid + 256]; } __syncthreads();
    }
    if (blockSize >= 256) {
        if (tid < 128) { sdata[tid] += sdata[tid + 128]; } __syncthreads();
    }
    if (blockSize >= 128) {
        if (tid < 64) { sdata[tid] += sdata[tid + 64]; } __syncthreads();
    }

    // Efficient warp reduce function
    if (tid < 32) warpReduce<blockSize>(sdata, tid);

    // Write the final result to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}