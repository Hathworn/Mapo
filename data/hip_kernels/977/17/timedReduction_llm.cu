#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void timedReduction(const float *input, float *output, clock_t *timer) {
    extern __shared__ float shared[];

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int offset = blockDim.x * 2 * bid;

    // Start timing
    if (tid == 0) timer[bid] = clock();

    // Copy input to shared memory efficiently
    float val = (tid + offset < gridDim.x * blockDim.x * 2) ? input[tid + offset] : FLT_MAX;
    float val2 = (tid + offset + blockDim.x < gridDim.x * blockDim.x * 2) ? input[tid + offset + blockDim.x] : FLT_MAX;
    shared[tid] = min(val, val2);

    // Perform reduction to find minimum
    for (int d = blockDim.x / 2; d > 0; d >>= 1) {
        __syncthreads();
        if (tid < d) {
            shared[tid] = min(shared[tid], shared[tid + d]);
        }
    }

    // Write result for this block
    if (tid == 0) output[bid] = shared[0];

    // End timing
    __syncthreads();
    if (tid == 0) timer[bid + gridDim.x] = clock();
}