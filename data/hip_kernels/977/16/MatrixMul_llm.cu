#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMul(int *M, int *N, int *P, int width)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    const int tile_size = 16;

    __shared__ int As[tile_size][tile_size];
    __shared__ int Bs[tile_size][tile_size];

    int aBegin = width * tile_size * by;
    int aEnd   = aBegin + width - 1;
    int aStep  = tile_size;

    int bBegin = tile_size * bx;
    int bStep  = tile_size * width;

    int Csub = 0;

    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep)
    {
        // Improved memory coalescing by maintaining proper stride
        As[ty][tx] = M[a + width * ty + tx];
        Bs[ty][tx] = N[b + width * ty + tx];
        __syncthreads();

        // No Bank Conflict and improved readability
        for (int k = 0; k < tile_size; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }
        __syncthreads();
    }

    int c = width * tile_size * by + tile_size * bx;
    P[c + width * ty + tx] = Csub;
}