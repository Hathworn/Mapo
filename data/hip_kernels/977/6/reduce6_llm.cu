#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

template <unsigned int blockSize>
__device__ void warpReduce(volatile int* sdata, int tid) {
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32];
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16];
    if (blockSize >= 16) sdata[tid] += sdata[tid + 8];
    if (blockSize >= 8) sdata[tid] += sdata[tid + 4];
    if (blockSize >= 4) sdata[tid] += sdata[tid + 2];
    if (blockSize >= 2) sdata[tid] += sdata[tid + 1];
}

template <unsigned int blockSize>
__global__ void reduce6(int *g_idata, int *g_odata, int n) {
    extern __shared__ int sdata[];

    // Calculate thread and grid indices
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (blockSize * 2) + threadIdx.x;
    unsigned int gridSize = blockSize * 2 * gridDim.x;
    
    // Initialize shared memory to zero
    int sum = 0;
    while (i < n) {
        // Perform first level of reduction in registers
        sum += g_idata[i] + g_idata[i + blockSize];
        i += gridSize; // Move to the next portion
    }
    sdata[tid] = sum; // Store thread's reduce result in shared memory
    __syncthreads();

    // Unrolled reduction in shared memory with syncthreads barrier
    if (blockSize >= 512) { if (tid < 256) sdata[tid] += sdata[tid + 256]; __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) sdata[tid] += sdata[tid + 128]; __syncthreads(); }
    if (blockSize >= 128) { if (tid < 64) sdata[tid] += sdata[tid + 64]; __syncthreads(); }
    
    // Warp reduce for the last 32 threads
    if (tid < 32) warpReduce<blockSize>(sdata, tid);

    // Write the block's result to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}