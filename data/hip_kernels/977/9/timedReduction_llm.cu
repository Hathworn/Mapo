#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void timedReduction(const float *input, float *output, clock_t *timer)
{
    extern __shared__ float shared[];

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    if (tid == 0) {
        // Initialize timer at the start
        timer[bid] = clock();
    }
    
    // Copy input to shared memory efficiently
    int index = bid * (2 * blockDim.x) + tid;
    shared[tid] = input[index];
    shared[tid + blockDim.x] = input[index + blockDim.x];

    // Perform reduction using shared memory
    for (int d = blockDim.x; d > 0; d >>= 1)
    {
        __syncthreads();
        
        if (tid < d)
        {
            float f0 = shared[tid];
            float f1 = shared[tid + d];
            shared[tid] = fminf(f0, f1);  // More concise operation
        }
    }

    // Write result
    if (tid == 0) {
        output[bid] = shared[0];

        // Record timer at the end
        timer[bid + gridDim.x] = clock();
    }
}