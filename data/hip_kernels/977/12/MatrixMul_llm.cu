#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMul(int *M, int *N, int *P, int width)
{
    int bx = blockIdx.x;
    int by = blockIdx.y;

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    const int tile_size = 16; // tile size

    __shared__ int As[tile_size][tile_size];
    __shared__ int Bs[tile_size][tile_size];

    int aBegin = width * tile_size * by;
    int bBegin = tile_size * bx;

    int Csub = 0;

    // Loop over tiles
    for (int a = aBegin, b = bBegin; a < aBegin + width; a += tile_size, b += tile_size * width)
    {
        // Collaborative loading of tiles into shared memory
        As[ty][tx] = M[a + width * ty + tx];
        Bs[ty][tx] = N[b + width * ty + tx];
        __syncthreads();

        // Compute dot product for one element of the C matrix
        for (int k = 0; k < tile_size; ++k)
        {
            Csub += As[ty][k] * Bs[k][tx];
        }
        __syncthreads();
    }

    // Write the block sub-matrix to global memory
    int c = width * tile_size * by + tile_size * bx;
    P[c + width * ty + tx] = Csub;
}