#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduce_sum_kernel(const float *input, float *d_out, int size) {
    int tid = threadIdx.x; // Local thread index
    int myId = blockIdx.x * (blockDim.x * 2) + threadIdx.x; // Global thread index

    extern __shared__ float tempsum[]; // shared memory

    // Load data into shared memory
    if (myId + blockDim.x < size) {
        tempsum[tid] = input[myId] + input[myId + blockDim.x];
    } else if (myId < size) {
        tempsum[tid] = input[myId];
    } else {
        tempsum[tid] = 0.0f;
    }

    __syncthreads(); // Ensure all shared memory loads are completed

    // Perform reduction using shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            tempsum[tid] += tempsum[tid + s];
        }
        __syncthreads(); // Ensure all memory operations are completed
    }

    // Write the result of block reduction to output
    if (tid == 0) {
        d_out[blockIdx.x] = tempsum[0];
    }
}