#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void pack(const int* prefix_sum, const int* src, int* dst, const int nb_vert) 
{
    // Use shared memory for faster access if possible
    __shared__ int shared_src[1024];
    __shared__ int shared_prefix_sum[1024];

    const int p = blockIdx.x * blockDim.x + threadIdx.x;

    if (p < nb_vert) {
        // Load data into shared memory
        shared_src[threadIdx.x] = src[p];
        shared_prefix_sum[threadIdx.x] = prefix_sum[p];
    }
    __syncthreads(); // Synchronize to ensure data is loaded in shared memory

    if (p < nb_vert) {
        const int elt = shared_src[threadIdx.x];
        if (elt >= 0) {
            dst[shared_prefix_sum[threadIdx.x]] = elt;
        }
    }
}