#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void transform_vert_to_fit(const int* __restrict__ src, int* __restrict__ dst, const int nb_vert)
{
    // Calculate the global index
    const int p = blockIdx.x * blockDim.x + threadIdx.x;

    // Efficient warp-level handling of elements
    if(p < nb_vert)
    {
        int src_value = src[p];
        dst[p] = (src_value >> 31) + 1; // Use bitwise operation for conversion
    }
}