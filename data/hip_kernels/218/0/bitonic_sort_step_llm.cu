#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bitonic_sort_step(float *dev_values, int j, int k)
{
    unsigned int i, ixj;
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i ^ j;

    // Only threads with valid partners proceed
    if (ixj > i)
    {
        float val_i = dev_values[i];
        float val_ixj = dev_values[ixj];

        // Determine direction of sorting and perform it
        bool ascending = ((i & k) == 0);
        if ((ascending && val_i > val_ixj) || (!ascending && val_i < val_ixj))
        {
            // Swap values using a temporary variable
            dev_values[i] = val_ixj;
            dev_values[ixj] = val_i;
        }
    }
}