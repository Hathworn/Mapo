#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_WindowGate2_updateOutput_kernel( float *output, float *centroids, float *normalizedCentroids, float *inputIndice, float *outputIndice, const float *input, const float *noise, int inputSize, int outputSize, int inputWindowSize, int outputWindowSize, int windowStride, int train)
{
    __shared__ float buffer[WINDOWGATE2_THREADS+1];
    unsigned int tx = threadIdx.x;
    unsigned int k = blockIdx.x;
    const float *input_k = input + inputSize*k;
    float *output_k = output + outputWindowSize*k;

    // Initialize shared memory
    buffer[tx] = 0.0f;
    __syncthreads();

    // Load input and compute the weighted sum
    float partialSum = 0.0f;
    for (unsigned int i = tx; i < inputSize; i += blockDim.x) {
        partialSum += input_k[i] * (float)(i + 1);
    }
    buffer[tx] = partialSum;
    __syncthreads();

    // Reduce using binary tree in shared memory
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        if (tx < stride) {
            buffer[tx] += buffer[tx + stride];
        }
        __syncthreads();
    }

    if (tx == 0) {
        float centroid = buffer[0];

        // Normalize centroid between 0 and 1
        centroid /= (float)(inputSize);
        normalizedCentroids[k] = centroid;
        if (train) {
            centroid += noise[k];
            centroid = fminf(fmaxf(0.0f, centroid), 1.0f);
        }

        // Align centroids to output
        centroid *= (float)(outputSize);

        // Calculate and clip indices
        float inputIdx = centroid / (float)(inputSize) - 0.5f * (float)inputWindowSize;
        float outputIdx = centroid - 0.5f * (float)outputWindowSize;
        inputIdx = fminf(inputIdx, (float)(inputSize - inputWindowSize + 1));
        inputIdx = fmaxf(inputIdx, 1.0f);
        outputIdx = fminf(outputIdx, (float)(outputSize - outputWindowSize + 1));
        outputIdx = fmaxf(outputIdx, 1.0f);

        // Align centroid to output window
        inputIdx = ceilf(inputIdx);
        outputIdx = ceilf(outputIdx);
        centroid -= (outputIdx - 1.0f);

        // Store inputs and outputs indices
        inputIndice[k] = (int)inputIdx;
        outputIndice[k] = (int)outputIdx;
        centroids[k] = centroid;

        // Store input index for later use
        buffer[WINDOWGATE2_THREADS] = inputIdx;
    }

    __syncthreads();

    // Load input window using calculated input index
    float inputIdx = buffer[WINDOWGATE2_THREADS];
    const float *inputWindow = input_k + (int)inputIdx;

    // Copy data to output window
    for (int i = tx; i < outputWindowSize; i += blockDim.x) {
        output_k[i] = inputWindow[(int)(floorf((float)i / windowStride))];
    }
}