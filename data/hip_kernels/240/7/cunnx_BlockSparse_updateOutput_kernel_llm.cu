#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_BlockSparse_updateOutput_kernel(float *output, const float *input, const float *outputIndice, const float *outputScale, const float *bias, int outputSize, int nOutputBlock, int inputWindowSize, int outputWindowSize)
{
    __shared__ float buffer[BLOCKSPARSE_THREADS];
    int tx = threadIdx.x;
    int k = blockIdx.x;
    
    float *output_k = output + k * outputWindowSize * outputSize;
    const float *input_k = input + k * inputWindowSize * outputWindowSize * outputSize;
    const float *outputIndice_k = outputIndice + k * outputWindowSize;
    const float *outputScale_k = outputScale + k * outputWindowSize;

    // Loop unrolling for improved performance
    for (int m = 0; m < outputWindowSize; ++m)
    {
        int outputIdx = static_cast<int>(outputIndice_k[m]) - 1;
        float outputScale = outputScale_k[m];

        // Allocate a separate buffer register for each thread
        float buffer_temp = 0.0f;

        for (int j = tx; j < outputSize; j += blockDim.x)
        {
            buffer_temp = bias[outputIdx * outputSize + j];

            #pragma unroll 4 // Unroll inner loop for better performance
            for (int l = 0; l < inputWindowSize; ++l)
            {
                buffer_temp += input_k[l * outputWindowSize * outputSize + m * outputSize + j];
            }

            output_k[m * outputSize + j] = outputScale * buffer_temp;
        }
    }
}