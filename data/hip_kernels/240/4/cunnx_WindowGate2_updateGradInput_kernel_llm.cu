#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_WindowGate2_updateGradInput_kernel( float *gradInput, float *error, float* targetCentroids, const float *centroids,const float *input, const float *inputIndice, const float *outputIndice, const float* output, const float* gradOutput, int inputSize, int outputSize, int inputWindowSize, int outputWindowSize, int windowStride, float c, float d, float e, float lr)
{
    // Cache common indices to reduce redundant computation
    unsigned int tx = threadIdx.x;
    unsigned int k = blockIdx.x;
    const float *gradOutput_k = gradOutput + outputWindowSize * k;
    float *gradInput_k = gradInput + inputSize * k;
    float *gradInputWindow = gradInput_k + (int)(inputIndice[k] - 1);

    // Use shared memory to reduce global memory access latency
    extern __shared__ float sharedGradOutput[];
    for (int i = tx; i < outputWindowSize; i += blockDim.x) {
        sharedGradOutput[i] = gradOutput_k[i];
    }
    __syncthreads();

    // Optimize the loop to reduce redundant calculations
    for (int i = tx; i < inputWindowSize; i += blockDim.x) {
        float sum = 0;
        for (int j = 0; j < windowStride; j++) {
            sum += sharedGradOutput[i * windowStride + j];
        }
        gradInputWindow[i] += sum;
    }
}