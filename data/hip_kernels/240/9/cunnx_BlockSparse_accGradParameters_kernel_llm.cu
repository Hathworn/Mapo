#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCKSPARSE_THREADS 256
#define BLOCKSPARSE_MAXOUTPUTBLOCKSIZE 128

__global__ void cunnx_BlockSparse_accGradParameters_kernel( float *gradWeight, float* gradBias, float *gradOutput, float *input, float *inputIndice, float *outputIndice, int inputSize, int outputSize, int nInputBlock, int nOutputBlock, int inputWindowSize, int outputWindowSize, float scale)
{
    __shared__ float buffer[BLOCKSPARSE_THREADS];
    __shared__ float gradOutputBuffer[BLOCKSPARSE_MAXOUTPUTBLOCKSIZE];
    int tx = threadIdx.x;
    int i_step = blockDim.x;
    int k = blockIdx.x;

    float *input_k = input + k * inputWindowSize * inputSize;
    float *gradOutput_k = gradOutput + k * outputWindowSize * outputSize;
    float *inputIndice_k = inputIndice + k * inputWindowSize;
    float *outputIndice_k = outputIndice + k * outputWindowSize;

    for (int m = 0; m < outputWindowSize; m++)
    {
        int outputIdx = (int)outputIndice_k[m] - 1;
        float *blockGradOutput = gradOutput_k + m * outputSize;
        float *blockGradBias = gradBias + outputIdx * outputSize;

        for (int j = tx; j < outputSize; j += i_step)
            gradOutputBuffer[j] = blockGradOutput[j] * scale;

        __syncthreads();

        for (int l = 0; l < inputWindowSize; l++)
        {
            int inputIdx = (int)inputIndice_k[l] - 1;
            float *blockInput = input_k + l * inputSize;
            float *blockGradWeight = gradWeight + outputIdx * nInputBlock * outputSize * inputSize + inputIdx * outputSize * inputSize;

            // Efficient global memory access by coalescing
            for (int i = tx; i < inputSize; i += i_step)
            {
                buffer[tx] = blockInput[i]; // Use shared memory for input

                // Optimized atomic operation
                for (int j = 0; j < outputSize; j++)
                    atomicAdd(&(blockGradWeight[j * inputSize + i]), gradOutputBuffer[j] * buffer[tx]);
            }
        }

        __syncthreads();

        // Efficient atomic addition for biases
        for (int j = tx; j < outputSize; j += i_step)
            atomicAdd(&(blockGradBias[j]), gradOutputBuffer[j]);
    }
}