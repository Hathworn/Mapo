#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunnx_LazyKBest_updateGradInput_kernel(float *gradInput, const float *indice, const float *gradOutput, int inputSize, int outputSize)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Compute global index

    if (i < outputSize) // Ensure within bounds
    {
        int k = blockIdx.x;
        float *gradInput_k = gradInput + k * inputSize;
        const float *gradOutput_k = gradOutput + k * outputSize;
        const float *indice_k = indice + k * outputSize;

        gradInput_k[(int)(indice_k[i] - 1)] = gradOutput_k[i]; // Assign computed value
    }
}