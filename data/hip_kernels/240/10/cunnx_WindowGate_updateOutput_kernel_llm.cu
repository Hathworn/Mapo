#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunnx_WindowGate_updateOutput_kernel(float *output, float *centroids, float *normalizedCentroids, float *outputIndice, const float *input, const float *noise, int inputSize, int outputSize, int outputWindowSize, float a, float b, int train)
{
    __shared__ float buffer[WINDOWGATE_THREADS];
    unsigned int tx = threadIdx.x;
    unsigned int k = blockIdx.x;
    const float *input_k = input + inputSize * k;
    float *output_k = output + outputWindowSize * k;

    // Initialize shared memory buffer to 0
    buffer[tx] = 0;

    // Calculate the weighted sum for centroid coordinate
    for (unsigned int i = tx; i < inputSize; i += blockDim.x)
        buffer[tx] += input_k[i] * (float)(i + 1);

    // Reduction to get single centroid value per block
    for (unsigned int stride = blockDim.x >> 1; stride > 0; stride >>= 1)
    {
        __syncthreads();
        if (tx < stride)
            buffer[tx] += buffer[tx + stride];
    }

    if (tx == 0)
    {
        float centroid = buffer[0];
        centroid /= (float)(inputSize); // Normalize centroid
        normalizedCentroids[k] = centroid;
        
        if (train)
        {
            centroid += noise[k];
            centroid = fminf(fmaxf(0, centroid), 1);
        }
        centroid *= (float)(outputSize); // Align centroid

        float outputIdx = centroid - 0.5f * (float)outputWindowSize;
        outputIdx = fminf(fmaxf(outputIdx, 1), outputSize - outputWindowSize + 1);
        outputIdx = ceilf(outputIdx); // Calculate output index
        
        centroid -= (outputIdx - 1); // Adjust centroid position

        outputIndice[k] = (int)outputIdx;
        centroids[k] = centroid;
        buffer[0] = centroid;
    }

    __syncthreads();
    float centroid = buffer[0];

    // Apply Gaussian blur to output window
    for (int i = tx; i < outputWindowSize; i += blockDim.x)
    {
        float x = (float)(i + 1) - centroid;
        output_k[i] = a * expf(x * x * b);
    }
}