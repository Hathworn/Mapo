#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void copy_fill(float *in, float *out, int size, int in_size2, int in_size3, int out_size2, int out_size3)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        // Compute output coordinates
        int out_x = id % out_size3;
        int out_y = id / out_size3;

        // Compute input coordinates
        int in_x = out_x - (out_size3 - in_size3) / 2;
        int in_y = out_y - (out_size2 - in_size2) / 2;

        // Clamp input coordinates within bounds
        int x = (in_x < 0) ? 0 : ((in_x >= in_size3) ? in_size3 - 1 : in_x);
        int y = (in_y < 0) ? 0 : ((in_y >= in_size2) ? in_size2 - 1 : in_y);

        // Perform the copy
        out[id] = in[y * in_size3 + x];
    }
}