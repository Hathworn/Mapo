#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_SpatialLogSoftMax_updateGradInput_kernel(float *gradInput, float *output, float *gradOutput, int feature_size, int spatial_size, int data_size)
{
    int idx = (threadIdx.x + blockDim.x * blockIdx.x);
    idx = (idx / spatial_size) * feature_size + idx % spatial_size;

    // Improved bounds check to prevent unnecessary computation
    if (idx >= data_size) return; 

    float gradSum = 0.0;
    int next_idx = idx + feature_size;

    // Unrolling loop for performance
    for (int i = idx; i < next_idx; i += spatial_size) {
        gradSum += gradOutput[i];
    }

    // Computing the new gradient
    for (int i = idx; i < next_idx; i += spatial_size) {
        gradInput[i] = gradOutput[i] - __expf(output[i]) * gradSum;
    }
}