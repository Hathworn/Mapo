#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TB 128

#define DISP_MAX 256

__global__ void subpixel_enchancement(float *d0, float *c2, float *out, int size, int dim23, int disp_max) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit if id is out of bounds
    if (id >= size) return;

    // Pre-calculated index for c2 array
    int d = d0[id];
    int baseDim23 = d * dim23 + id;

    out[id] = d;

    // Check valid disparity range
    if (1 <= d && d < disp_max - 1) {
        // Load neighboring values of c2
        float cn = c2[(d - 1) * dim23 + id];
        float cz = c2[baseDim23];
        float cp = c2[(d + 1) * dim23 + id];

        // Calculate denominator and check its validity
        float denom = 2 * (cp + cn - 2 * cz);
        if (denom > 1e-5) {
            // Compute sub-pixel enhancement
            out[id] = d - min(1.0f, max(-1.0f, (cp - cn) / denom));
        }
    }
}