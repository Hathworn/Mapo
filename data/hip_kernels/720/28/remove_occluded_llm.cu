#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void remove_occluded(float *y, int size, int size3)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        int x = id % size3;
        float current_y = y[id]; // Cache the current y value
        for (int i = 1; x + i < size3; i++) {
            if (i - y[id + i] < -current_y) { // Use cached value
                y[id] = 0; // Mark as occluded
                break;
            }
        }
    }
}