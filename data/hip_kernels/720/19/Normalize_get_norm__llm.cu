#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void Normalize_get_norm_(float *input, float *norm, int size1, int size23, int size023)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size023) {
        int dim23 = id % size23;
        int dim0 = id / size23;

        float sum = 0.0f;
        float x;

        // Loop unrolling to improve performance
        int dim1;
        for (dim1 = 0; dim1 + 3 < size1; dim1 += 4) {
            x = input[(dim0 * size1 + dim1) * size23 + dim23];
            sum += x * x;
            x = input[(dim0 * size1 + dim1 + 1) * size23 + dim23];
            sum += x * x;
            x = input[(dim0 * size1 + dim1 + 2) * size23 + dim23];
            sum += x * x;
            x = input[(dim0 * size1 + dim1 + 3) * size23 + dim23];
            sum += x * x;
        }

        // Handle remaining elements
        for (; dim1 < size1; dim1++) {
            x = input[(dim0 * size1 + dim1) * size23 + dim23];
            sum += x * x;
        }

        norm[dim0 * size23 + dim23] = sum + 1e-5f;
    }
}