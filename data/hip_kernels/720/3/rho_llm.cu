#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void rho(float *x, int size, float lambda)
{
    // Calculate the global thread index.
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if thread index is within the array boundary.
    if (id < size) {
        // Perform computation using shared memory.
        __shared__ float cached_lambda;
        if (threadIdx.x == 0) {
            cached_lambda = lambda;
        }
        __syncthreads(); // Ensure all threads have updated shared memory.

        // Optimize computation with shared data.
        x[id] = 1 - expf(-x[id] / cached_lambda);
    }
}