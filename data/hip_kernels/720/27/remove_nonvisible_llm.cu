#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void remove_nonvisible(float *y, int size, int size3)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < size) {
        int x = id % size3;
        float id_val = y[id];  // Load the value into a register for faster access
        if (id_val >= x) {
            y[id] = 0;
        }
    }
}