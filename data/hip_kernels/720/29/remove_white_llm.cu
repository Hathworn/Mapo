#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
extern "C" {
}

#define TB 128
#define DISP_MAX 256

__global__ void remove_white(float *x, float *y, int size)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return; // Early exit condition for out-of-bounds
    float value = x[id];
    y[id] = (value == 255) ? 0 : y[id]; // Use a ternary operator for brevity
}