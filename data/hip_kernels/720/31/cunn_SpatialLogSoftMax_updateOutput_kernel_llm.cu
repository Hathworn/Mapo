#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_SpatialLogSoftMax_updateOutput_kernel(float *output, float *input, int feature_size, int spatial_size, int data_size, float constant)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    idx = (idx / spatial_size) * feature_size + idx % spatial_size;

    if (idx < data_size) {
        int next_idx = idx + feature_size;
        float logsum = 0.0f;
        float max = -FLT_MAX; // Use predefined constant for min float value

        // Use shared memory to improve access speed
        __shared__ float s_input[1024]; // Adjust size if needed based on block size
        __shared__ float s_max;

        // Load input into shared memory
        for (int i = idx; i < next_idx; i += spatial_size) {
            if (threadIdx.x == 0) {
                s_max = max;
            }
            __syncthreads();

            // Find max in a parallel reduction
            
            atomicMax(&s_max, input[i]);

            __syncthreads();
            max = s_max; // Update max

            if (isnan(input[i])) continue;

            // Accumulate logsum using shared memory
            if (threadIdx.x == 0) {
                s_input[i % 1024] = __expf(input[i] - max);
            }
            atomicAdd(&logsum, s_input[i % 1024]);
        }

        logsum += constant;
        logsum = __logf(logsum) + max;

        // Compute logsoftmax
        for (int i = idx; i < next_idx; i += spatial_size) {
            output[i] = input[i] - logsum;
        }
    }
}