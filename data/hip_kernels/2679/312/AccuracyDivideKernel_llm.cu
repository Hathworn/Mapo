#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel to launch fewer threads, using a single atomic operation
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    if (threadIdx.x == 0) { // Ensure only one thread performs the division
        atomicExch(accuracy, *accuracy / N); // Use atomic operation to prevent race condition
    }
}