#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];

    // Load data into shared memory
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    tmp[threadIdx.x] = input[index];
    __syncthreads();

    // Perform unrolled reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            tmp[threadIdx.x] += tmp[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        output[blockIdx.x] = tmp[0];
    }
}