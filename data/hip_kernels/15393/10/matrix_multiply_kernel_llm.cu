#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    // Calculate the global thread index
    long i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread is within bounds
    if (i < dim_mn) {
        double out = 0.0;
        // Use shared memory for vector_in to reduce global memory access
        extern __shared__ double shared_vector[];
        
        // Load vector_in to shared memory
        if (threadIdx.x < dim_mn) {
            shared_vector[threadIdx.x] = vector_in[threadIdx.x];
        }
        __syncthreads();  // Synchronize threads in block to ensure complete loading
        
        // Perform matrix-vector multiplication
        for (long j = 0; j < dim_mn; j++) {
            out += matrix[i * dim_mn + j] * shared_vector[j];
        }
        
        // Write result to the output vector
        vector_out[i] = out;
    }
}