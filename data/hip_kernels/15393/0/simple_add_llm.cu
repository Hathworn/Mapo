#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_add(float* a, float* b, int n)
{
    // Get global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a stride loop to allow multiple active warps if data size is large
    int stride = blockDim.x * gridDim.x;
    
    for (; i < n; i += stride) {
        a[i] = a[i] + b[i];
    }
}