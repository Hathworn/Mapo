#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bigstencil(int* in, int* out) {
    // Use shared memory to reduce global memory accesses
    extern __shared__ int shared_in[];
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data from global to shared memory
    shared_in[threadIdx.x] = in[i];
    __syncthreads();

    // Perform computation
    out[i] = shared_in[threadIdx.x] + 2;
}