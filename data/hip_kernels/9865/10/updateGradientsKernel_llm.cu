#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void updateGradientsKernel(float4 *D, float4 *TD, unsigned int nVertices)
{
    int vidx = 4 * (blockIdx.x * blockDim.x) + threadIdx.x;
    if (vidx < nVertices) {
        // Directly copy 4 elements from TD to D
        D[vidx] = TD[vidx];
        if (vidx + BLOCK_SIZE_AVGG < nVertices) D[vidx + BLOCK_SIZE_AVGG] = TD[vidx + BLOCK_SIZE_AVGG];
        if (vidx + 2 * BLOCK_SIZE_AVGG < nVertices) D[vidx + 2 * BLOCK_SIZE_AVGG] = TD[vidx + 2 * BLOCK_SIZE_AVGG];
        if (vidx + 3 * BLOCK_SIZE_AVGG < nVertices) D[vidx + 3 * BLOCK_SIZE_AVGG] = TD[vidx + 3 * BLOCK_SIZE_AVGG];
    }
}