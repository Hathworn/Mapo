#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GradientAverageKernel(float4 *D, float4 *TD, unsigned int *NEIGHBOR, unsigned int *NBOFFSETS, unsigned int *nNeighbors, unsigned int nVertices)
{
    // Calculate starting index
    int vidxb = 4 * (blockIdx.x * blockDim.x) + threadIdx.x;

    // Shared memory for caching 4 elements per thread block
    __shared__ float4 SI[4 * BLOCK_SIZE_AVGG];
    
    // Thread index for accessing shared memory
    int bidx = 4 * threadIdx.x;
    
    // Cache vertices to shared memory
    for (int vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_AVGG; vidx += BLOCK_SIZE_AVGG) {
        if (vidx < nVertices) {
            SI[bidx] = D[vidx];
            bidx++;
        }
    }
    
    __syncthreads(); // Synchronize threads to ensure shared memory is fully populated
    
    bidx = 4 * threadIdx.x;
    
    // Process each vertex
    for (int vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_AVGG; vidx += BLOCK_SIZE_AVGG) {
        if (vidx < nVertices) {
            int offset = NBOFFSETS[vidx];
            int N = nNeighbors[vidx];
            float4 td = SI[bidx++];
            
            // Accumulate neighbor contributions
            for (int n = 0; n < N; n++) {
                int soffset = NEIGHBOR[offset + n];
                float4 nbd = D[soffset];
                
                td.x += nbd.x;
                td.y += nbd.y;
                td.z += nbd.z;
            }
            
            // Average and assign to output
            float invN = 1.0f / (float)(N + 1);
            td.x *= invN;
            td.y *= invN;
            td.z *= invN;
            
            TD[vidx] = td;
        }
    }
}