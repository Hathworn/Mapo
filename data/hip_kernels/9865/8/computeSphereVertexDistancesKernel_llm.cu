#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK_SIZE_CVD 128

__global__ void computeSphereVertexDistancesKernel(float4 *V, float *dist, unsigned int *NEIGHBOR, unsigned int *NBOFFSETS, unsigned int *nNeighbors, unsigned int nVertices, float circumference)
{
    int n, N;
    int offset, soffset;

    // Calculate global vertex index
    int vidxb = 4*(blockIdx.x * blockDim.x) + threadIdx.x;
    int basevert = 4*(blockIdx.x * blockDim.x);

    int vidx, tab;
    float4 nv, tv;
    float dot, n1, n2, norm;

    // Cache vertices in shared memory for block
    __shared__ float4 SI[4*BLOCK_SIZE_CVD];

    int bidx = threadIdx.x;
    // Load vertices into shared memory
    for (vidx = vidxb; vidx < vidxb + 4*BLOCK_SIZE_CVD; vidx += BLOCK_SIZE_CVD) {
        if (vidx < nVertices) {
            SI[bidx] = V[vidx];
            bidx += BLOCK_SIZE_CVD;
        }
    }

    // Ensure all threads complete loading
    __syncthreads();

    bidx = threadIdx.x;
    // Iterate over vertices for calculations
    for (vidx = vidxb; vidx < vidxb + 4*BLOCK_SIZE_CVD; vidx += BLOCK_SIZE_CVD) {
        if (vidx < nVertices) {
            offset = NBOFFSETS[vidx];
            N = nNeighbors[vidx];
            tv = SI[bidx];
            bidx += BLOCK_SIZE_CVD;

            for (n = 0; n < N; n++) {
                soffset = NEIGHBOR[offset + n];

                // Prefer shared memory when possible
                tab = soffset - basevert;
                nv = (tab >= 0 && tab < 4*BLOCK_SIZE_CVD) ? SI[tab] : V[soffset];

                // Use FMAD instructions for dot product calculation
                dot = __fmaf_rn(tv.x, nv.x, __fmaf_rn(tv.y, nv.y, __fmul_rn(tv.z, nv.z)));

                // Use FMAD instructions for norm calculations
                n1 = __fmaf_rn(tv.x, tv.x, __fmaf_rn(tv.y, tv.y, __fmul_rn(tv.z, tv.z)));
                n2 = __fmaf_rn(nv.x, nv.x, __fmaf_rn(nv.y, nv.y, __fmul_rn(nv.z, nv.z)));

                // Calculate norm using fast multiplication
                norm = __fmul_rn(__frsqrt_rn(n1), __frsqrt_rn(n2));

                // Efficient threshold checks and distance assignment
                if (norm >= 1.0e-7f) {
                    float scaled_dot = __fmul_rn(dot, __frcp_rn(norm));
                    dist[offset + n] = (fabsf(scaled_dot) <= 1.0f) ? __fmul_rn(circumference, fabsf(acosf(scaled_dot))) : 0.0f;
                } else {
                    dist[offset + n] = 0.0f;
                }
            }
        }
    }
}