#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuAddRNorm(float *dist, int width, int pitch, int height, float *vec) {
    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int xIndex = blockIdx.x * blockDim.x + tx;
    unsigned int yIndex = blockIdx.y * blockDim.y + ty;
    
    __shared__ float shared_vec[16];
    
    // Load vector values into shared memory with all threads in y-dimension
    if (tx == 0 && yIndex < height) {
        shared_vec[ty] = vec[yIndex];
    }
    
    __syncthreads();
    
    // Perform computation with valid indices
    if (xIndex < width && yIndex < height) {
        dist[yIndex * pitch + xIndex] += shared_vec[ty];
    }
}