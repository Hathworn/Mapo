#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuComputeNorm(float *mat, int width, int pitch, int height, float *norm) {
    unsigned int xIndex = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure valid thread work only if xIndex < width
    if (xIndex < width) {
        float sum = 0.0f;

        // Optimize loop by using strided access pattern
        for (int i = threadIdx.y; i < height; i += blockDim.y) {
            float val = mat[i * pitch + xIndex];
            sum += val * val;
        }

        // Use shared memory for intermediate results
        __shared__ float sharedSum[TILE_DIM];
        sharedSum[threadIdx.y] = sum;

        // Ensure all shared memory loads are done before reduction
        __syncthreads();

        // Perform reduction in shared memory
        if (threadIdx.y == 0) {
            float totalSum = 0.0f;
            for (int j = 0; j < blockDim.y; j++) {
                totalSum += sharedSum[j];
            }
            norm[xIndex] = totalSum;
        }
    }
}