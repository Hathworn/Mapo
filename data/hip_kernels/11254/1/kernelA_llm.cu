#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelA(float* r, float* x, float* y, float* z, int size) 
{
    // Use grid-stride loop for better occupancy and performance
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    for (; i < size; i += stride) {
        r[i] = x[i] * y[i] + z[i];
    }
}