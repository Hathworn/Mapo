#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use global index for processing
    if (index < size) {
        r[index] = x[index] * y[index] + z[index];
    }
}