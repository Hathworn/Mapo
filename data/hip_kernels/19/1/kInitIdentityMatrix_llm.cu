#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kInitIdentityMatrix(float* a, int size, int num_elements) {
    // Use a single index for row and column calculation
    const int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we don't access out of bounds
    if (x < num_elements) {
        // Calculate row and column and check if it is a diagonal element
        const int row = x / size;
        const int col = x % size;
        
        // Set matrix element to 1 if it's on the diagonal, otherwise 0
        a[x] = (row == col) ? 1.0f : 0.0f;
    }
}