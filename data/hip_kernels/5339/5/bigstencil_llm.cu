#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bigstencil(int* __restrict__ in, int* __restrict__ out) {
    // Use shared memory for reduced global memory access latency
    extern __shared__ int s_data[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Load input data into shared memory
    s_data[threadIdx.x] = in[i];
    __syncthreads();
    
    // Perform computation using shared memory
    out[i] = s_data[threadIdx.x] + 2;
}