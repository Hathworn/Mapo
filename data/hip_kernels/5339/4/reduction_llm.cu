#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];

    // Load data into shared memory
    int tid = threadIdx.x;
    int index = tid + blockIdx.x * blockDim.x;
    tmp[tid] = input[index];

    __syncthreads();

    // Perform reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            tmp[tid] += tmp[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to output
    if (tid == 0) {
        output[blockIdx.x] = tmp[0];
    }
}