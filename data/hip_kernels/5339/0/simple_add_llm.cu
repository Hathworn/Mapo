#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_add(float* __restrict__ a, const float* __restrict__ b, int n)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if(i < n) {
        // Perform addition
        a[i] += b[i];
    }
}