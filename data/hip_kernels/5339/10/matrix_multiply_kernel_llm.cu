#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    // Utilizing shared memory to optimize data access
    extern __shared__ double shared_vector_in[];
    long i = threadIdx.x + blockIdx.x * blockDim.x;
    double out = 0.0;

    if (threadIdx.x < dim_mn) {
        shared_vector_in[threadIdx.x] = vector_in[threadIdx.x];
    }
    __syncthreads();

    if (i < dim_mn) {
        for (long j = 0; j < dim_mn; j++) {
            out += matrix[i * dim_mn + j] * shared_vector_in[j];
        }
        vector_out[i] = out;
    }
}