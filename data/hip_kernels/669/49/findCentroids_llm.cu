#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findCentroids(int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount) {
    __shared__ int localMasses[32*17];
    __shared__ unsigned int localCounts[32];
    int pixel = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadIdx.x < 32) {
        // Initialize shared memory
        for (int i = 0; i < 17; i++) {
            localMasses[32 * i + threadIdx.x] = 0;
        }
        localCounts[threadIdx.x] = 0;
    }
    __syncthreads();

    if (pixel < nPixels) {
        int myCluster = cluster[pixel];
        int myIndex = pixel;
        // Accumulate local masses
        for(int filter = 0; filter < 17; filter++) {
            int myElement = responses[myIndex];
            atomicAdd(localMasses + filter * 32 + myCluster, myElement);
            myIndex += nPixels;
        }
        // Increment local counts
        atomicAdd(&localCounts[myCluster], 1);
    }
    __syncthreads();

    if (threadIdx.x < 32) {
        // Update global centroid masses
        for (int filter = 0; filter < 17; filter++) {
            atomicAdd(centroidMass + filter * 32 + threadIdx.x, localMasses[threadIdx.x + filter * 32]);
        }
        // Reset shared memory for second phase
        for (int i = 0; i < 17; i++) {
            localMasses[32 * i + threadIdx.x] = 0;
        }
    }
    __syncthreads();

    if (pixel < nPixels) {
        int myCluster = cluster[pixel];
        int myIndex = pixel + nPixels * 17;
        // Accumulate additional responses
        for(int filter = 0; filter < 17; filter++) {
            int myElement = responses[myIndex];
            atomicAdd(localMasses + filter * 32 + myCluster, myElement);
            myIndex += nPixels;
        }
    }
    __syncthreads();

    if (threadIdx.x < 32) {
        // Update global centroid masses with second set
        for (int filter = 17; filter < 34; filter++) {
            atomicAdd(centroidMass + filter * 32 + threadIdx.x, localMasses[threadIdx.x + (filter - 17) * 32]);
        }
        // Update global centroid counts
        atomicAdd(centroidCount + threadIdx.x, localCounts[threadIdx.x]);
    }
}