#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void assignInitialClusters(int width, int height, int nPixels, int clusterCount, int* cluster, int filterCount, float* responses, int* intResponses) {
    // Calculate global thread coordinates
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Calculate pixel index
    int pixel = y * width + x;

    // Check if within image bounds
    if ((x < width) && (y < height)) {
        // Precompute division factors
        int widthFactor = (width - 1) / 8 + 1;
        int heightFactor = (height - 1) / 8 + 1;
        
        // Optimization: Precompute division results
        int xBlock = x / widthFactor;
        int yBlock = y / heightFactor;
        
        // Assign cluster based on block coordinates
        int assignedCluster = yBlock * 8 + xBlock;
        cluster[pixel] = assignedCluster;
        
        // Conversion of responses in parallel manner
        for(int i = 0; i < filterCount; i++) {
            int index = pixel + i * nPixels;
            intResponses[index] = static_cast<int>(INTCONFACTOR * responses[index]);
        }
    }
}