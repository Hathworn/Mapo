#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findCentroidsAtomicFreeLocal_64(int afLocal, int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount)
{
    // Local variables to store the sum and count of responses for each thread
    int local_responses = 0;
    int local_count = 0;

    // Calculate pixel range for current af_id (blockIdx.x)
    int pixel_start = blockIdx.x * afLocal;
    int pixel_end = min((blockIdx.x + 1) * afLocal, nPixels);

    // Memory access pointers for current filter and cluster
    int* filter_responses = &responses[threadIdx.x * nPixels];
    
    // Iterate over assigned pixel range
    for (int i = pixel_start; i < pixel_end; i++)
    {
        if (cluster[i] == blockIdx.y)  // Check if pixel belongs to current cluster
        {
            local_responses += filter_responses[i];  // Accumulate response
            local_count++;  // Count the pixel
        }
    }

    // Calculate index for storing results using thread and block identifiers
    int idx = blockIdx.x * gridDim.y * blockDim.x + threadIdx.x * 64 + blockIdx.y;
    centroidMass[idx] = local_responses;  // Store local sum of responses
    centroidCount[idx] = local_count;  // Store local count of pixels
}