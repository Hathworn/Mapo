#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void assignInitialClusters(int width, int height, int nPixels, int clusterCount, int* cluster, int filterCount, float* responses, int* intResponses) {
    // Calculate unique thread index
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int pixel = y * width + x;
    
    // Ensure thread operates within image bounds
    if ((x < width) && (y < height)) {
        // Precompute block index values to reduce operations
        int xBlockFactor = ((width - 1) / 8 + 1);
        int yBlockFactor = ((height - 1) / 8 + 1);
        int xBlock = x / xBlockFactor;
        int yBlock = y / yBlockFactor;
        
        // Determine assigned cluster for the pixel
        int assignedCluster = yBlock * 8 + xBlock;
        cluster[pixel] = assignedCluster;

        // Process responses in batches using the pixel index
        int baseIndex = pixel;
        for(int i = 0; i < filterCount; i++, baseIndex += nPixels) {
            int response = (int)(INTCONFACTOR * responses[baseIndex]);
            intResponses[baseIndex] = response;
        }
    }
}