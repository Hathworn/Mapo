#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findCentroidsAtomicFreeReduce(int afLocal, int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount)
{
    int const af_id = blockIdx.x;
    int const cluster_id = blockIdx.y;
    int const filter_id = threadIdx.x;

    // Shared memory to accumulate local masses and counts
    __shared__ int smem_mass[64];
    __shared__ unsigned int smem_count[64];

    int local_mass = 0;
    int local_count = 0;

    if (af_id == 0)
    {
        int idx0 = filter_id*64 + cluster_id;

        for (int i=0; i<gridDim.x; i++)
        {
            int idxother = i * gridDim.y*blockDim.x + idx0;

            local_mass += centroidMass[idxother];
            local_count += centroidCount[idxother];
        }

        // Store results in shared memory
        smem_mass[filter_id] = local_mass;
        smem_count[filter_id] = local_count;
    }

    // Synchronize threads in the block
    __syncthreads();

    // Use a single thread to consolidate results in global memory
    if (filter_id == 0)
    {
        int total_mass = 0;
        unsigned int total_count = 0;

        for (int i = 0; i < 64; ++i) 
        {
            total_mass += smem_mass[i];
            total_count += smem_count[i];
        }

        centroidMass[cluster_id] = total_mass;
        centroidCount[cluster_id] = total_count;
    }
}