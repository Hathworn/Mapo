#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void findCentroidsAtomicFreeReduce(int afLocal, int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount)
{
    int const af_id = blockIdx.x;
    int const cluster_id = blockIdx.y;
    int const filter_id = threadIdx.x;
    
    extern __shared__ int shared_data[]; // Use shared memory for intermediate storage

    int* shared_mass = shared_data;
    int* shared_count = shared_data + blockDim.x * gridDim.y; // Ensure non-overlapping allocation

    shared_mass[filter_id * gridDim.y + cluster_id] = 0;
    shared_count[filter_id * gridDim.y + cluster_id] = 0;

    __syncthreads();

    // Only one thread needs to do this
    if (af_id == 0)
    {
        int idx0 = filter_id * 64 + cluster_id;
        
        for (int i = 0; i < gridDim.x; i++)
        {
            int idxother = i * gridDim.y * blockDim.x + idx0;
            
            atomicAdd(&shared_mass[filter_id * gridDim.y + cluster_id], centroidMass[idxother]);
            atomicAdd(&shared_count[filter_id * gridDim.y + cluster_id], centroidCount[idxother]);
        }

        __syncthreads(); // Ensure all atomic operations are complete

        // Write back results to global memory
        if (filter_id == 0)
        {
            centroidMass[idx0] = shared_mass[filter_id * gridDim.y + cluster_id];
            centroidCount[idx0] = shared_count[filter_id * gridDim.y + cluster_id];
        }
    }
}