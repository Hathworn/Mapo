#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void finishCentroids(int* centroidMass, unsigned int* centroidCount, float* centroids) {
    // Flatten thread and block indices for 2D grid
    int centroidNumber = blockIdx.y * blockDim.y + threadIdx.y;
    int dimensionNumber = blockIdx.x * blockDim.x + threadIdx.x;
    int index = dimensionNumber * 64 + centroidNumber; // Calculate index once

    // Ensure within bounds
    if (centroidNumber < 64 && dimensionNumber < 34) {
        float totalCount = (float)centroidCount[centroidNumber];
        float mass = (float)centroidMass[index];
        centroids[index] = mass / ((float)INTCONFACTOR * totalCount);
    }
}