#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void quantizeImage_kernel(uint width, uint height, uint nbins, float* devInput, int* devOutput) {
    // Use shared memory to reduce global memory access
    extern __shared__ float sharedInput[];

    int x0 = blockDim.x * blockIdx.x + threadIdx.x;
    int y0 = blockDim.y * blockIdx.y + threadIdx.y;

    // Load input into shared memory
    if ((x0 < width) && (y0 < height)) {
        int index = y0 * width + x0;
        sharedInput[threadIdx.y * blockDim.x + threadIdx.x] = devInput[index];
    }
    __syncthreads();

    // Perform quantization using shared memory
    if ((x0 < width) && (y0 < height)) {
        int index = y0 * width + x0;
        float input = sharedInput[threadIdx.y * blockDim.x + threadIdx.x];
        int output = (int)floorf(input * (float)nbins);
        
        // Optimize the output calculation
        devOutput[index] = min(output, nbins - 1);
    }
}