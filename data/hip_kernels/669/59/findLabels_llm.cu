#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findLabels(int nPixels, int filterCount, int clusterCount, float* responses, float* centroids, int* clusters, int* changes) {
    __shared__ float sharedCentroids[34 * 32];
    __shared__ unsigned int localChanges;
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    // Load the first set of 32 centroids into shared memory
    if (threadIdx.x < 32) {
        for (int i = 0; i < 34; i++) {
            sharedCentroids[i * 32 + threadIdx.x] = centroids[i * 64 + threadIdx.x];
        }
    }
    __syncthreads();

    int bestLabel = -1;
    float bestDistance = 1000000.0f;

    // Calculate distances for the first set of 32 centroids
    if (x < nPixels) {
        for (int label = 0; label < 32; label++) {
            float accumulant = 0.0f;
            int index = x;
            for (int dimension = 0; dimension < 34; dimension++) {
                float diff = sharedCentroids[dimension * 32 + label] - responses[index];
                accumulant += diff * diff;
                index += nPixels;
            }
            if (accumulant < bestDistance) {
                bestLabel = label;
                bestDistance = accumulant;
            }
        }
    }

    // Load the second set of 32 centroids into shared memory
    if (threadIdx.x < 32) {
        for (int i = 0; i < 34; i++) {
            sharedCentroids[i * 32 + threadIdx.x] = centroids[i * 64 + threadIdx.x + 32];
        }
    }
    __syncthreads();

    // Calculate distances for the second set of 32 centroids
    if (x < nPixels) {
        for (int label = 0; label < 32; label++) {
            float accumulant = 0.0f;
            int index = x;
            for (int dimension = 0; dimension < 34; dimension++) {
                float diff = sharedCentroids[dimension * 32 + label] - responses[index];
                accumulant += diff * diff;
                index += nPixels;
            }
            if (accumulant < bestDistance) {
                bestLabel = label + 32;
                bestDistance = accumulant;
            }
        }
        // Update cluster changes
        int formerCluster = clusters[x];
        if (bestLabel != formerCluster) {
            atomicAdd(&localChanges, 1); // Use atomicAdd for changes
            clusters[x] = bestLabel;
        }
    }
    __syncthreads();

    // Accumulate total changes
    if (threadIdx.x == 0) {
        atomicAdd(changes, localChanges);
    }
}