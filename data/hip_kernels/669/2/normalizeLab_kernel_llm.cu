#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalizeLab_kernel(uint width, uint height, float* devL, float* devA, float* devB) {
    // Compute global thread indices
    int x0 = blockDim.x * blockIdx.x + threadIdx.x;
    int y0 = blockDim.y * blockIdx.y + threadIdx.y;

    // Check if within image bounds
    if ((x0 < width) && (y0 < height)) {
        int index = y0 * width + x0;

        const float ab_min = -73;
        const float ab_max = 95;
        const float ab_range = ab_max - ab_min;

        // Normalize Lab image
        float l_val = fminf(fmaxf(devL[index] / 100.0f, 0.0f), 1.0f);
        float a_val = fminf(fmaxf((devA[index] - ab_min) / ab_range, 0.0f), 1.0f);
        float b_val = fminf(fmaxf((devB[index] - ab_min) / ab_range, 0.0f), 1.0f);

        devL[index] = l_val;
        devA[index] = a_val;
        devB[index] = b_val;
    }
}