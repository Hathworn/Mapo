#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fillImage(int width, int height, int value, int* devOutput) {
    // Each thread calculates its own index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate the index only if the coordinates are within bounds
    if (x < width && y < height) {
        int index = y * width + x; // moved inside boundary check to reduce unnecessary calculations
        devOutput[index] = value;  // Set the pixel value
    }
}