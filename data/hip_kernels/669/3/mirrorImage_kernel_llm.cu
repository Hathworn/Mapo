#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mirrorImage_kernel(uint width, uint height, uint border, uint borderWidth, uint borderHeight, float* devInput, float* devOutput) {
    int x0 = blockDim.x * blockIdx.x + threadIdx.x;
    int y0 = blockDim.y * blockIdx.y + threadIdx.y;

    // Limit processing to threads that fall within the border region
    if ((x0 < borderWidth) && (y0 < borderHeight)) {
        int x1, y1;

        // Calculate mirrored x-coordinate
        if (x0 < border) {
            x1 = border - x0 - 1;
        } else if (x0 >= border + width) {
            x1 = border + 2 * width - x0 - 1;
        } else {
            x1 = x0 - border;
        }

        // Calculate mirrored y-coordinate
        if (y0 < border) {
            y1 = border - y0 - 1;
        } else if (y0 >= border + height) {
            y1 = border + 2 * height - y0 - 1;
        } else {
            y1 = y0 - border;
        }

        // Assign the mirrored pixel value
        devOutput[y0 * borderWidth + x0] = devInput[y1 * width + x1];
    }
}