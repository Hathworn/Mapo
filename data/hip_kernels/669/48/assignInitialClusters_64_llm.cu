#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void assignInitialClusters_64(int width, int height, int nPixels, int clusterCount, int* cluster, int filterCount, float* responses, int* intResponses) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    int pixel = y * width + x;

    // Check bounds
    if ((x < width) && (y < height)) {
        // Use more efficient division
        int xBlock = __float2int_rd((float)x * 8.0f / width);
        int yBlock = __float2int_rd((float)y * 8.0f / height); 
        int assignedCluster = yBlock * 8 + xBlock;

        cluster[pixel] = assignedCluster;

        // Unroll the loop for better performance
        for (int i = 0; i < filterCount; i++) {
            int index = pixel + i * nPixels;
            int response = __fmul_rd(INTCONFACTOR, responses[index]);
            intResponses[index] = response;
        }
    }
}