#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findDiffLabels(float* devDiff, int diffPitchInFloats, int nPoints, int nClusters, int* devClusters, int* devChanges) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ unsigned int localChanges;
    
    // Initialize shared variable only once per block
    if (threadIdx.x == 0) {
        localChanges = 0;
    }
    __syncthreads();

    if (x < nPoints) {
        int index = x;
        float minDistance = FLT_MAX; // Use FLT_MAX for clearer intent
        int minCluster = -1;

        for(int cluster = 0; cluster < nClusters; cluster++) {
            float clusterDistance = devDiff[index];

            if (clusterDistance < minDistance) {
                minDistance = clusterDistance;
                minCluster = cluster;
            }
            index += diffPitchInFloats;
        }
        
        int previousCluster = devClusters[x];
        devClusters[x] = minCluster;

        // Use atomicAdd for changes, more efficient in context
        if (minCluster != previousCluster) {
            atomicAdd(&localChanges, 1);
        }
    }
    __syncthreads();

    // Reduce local changes from all threads to a single global change
    if (threadIdx.x == 0) {
        atomicAdd(devChanges, localChanges);
    }
}