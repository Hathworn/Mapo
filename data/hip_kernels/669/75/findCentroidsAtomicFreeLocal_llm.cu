#include "hip/hip_runtime.h"
```cuda
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findCentroidsAtomicFreeLocal(int afLocal, int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount)
{
    int const af_id = blockIdx.x;
    int const cluster_id = blockIdx.y;
    int const filter_id = threadIdx.x;
    int* filter_responses = &responses[filter_id * nPixels];

    // Use shared memory to reduce global memory access
    __shared__ int shared_responses[256];
    __shared__ int shared_counts[256];

    shared_responses[filter_id] = 0;
    shared_counts[filter_id] = 0;

    int pixel_start = af_id * afLocal;
    int pixel_end = min((af_id + 1) * afLocal, nPixels);

    for (int i = pixel_start; i < pixel_end; i++) {
        if (cluster[i] == cluster_id) {
            shared_responses[filter_id] += filter_responses[i];
            shared_counts[filter_id]++;
        }
    }

    // Synchronize threads within block to ensure shared memory is updated
    __syncthreads();

    // Write accumulated results back to global memory
    int idx = af_id * gridDim.y * blockDim.x + filter_id * 64 + cluster_id;
    centroidMass[idx] = shared_responses[filter_id];
    centroidCount[idx] = shared_counts[filter_id];
}