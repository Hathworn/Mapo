#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMaxMinPerGrid(int p_nGridSize, int p_nEigNum, float* p_devMax, float* p_devMin, float* p_devReduceMax, float* p_devReduceMin, int p_nMaxLevel)
{
    __shared__ float MaxReduce[XBLOCK*(MAXEIGNUM - 1)];
    __shared__ float MinReduce[XBLOCK*(MAXEIGNUM - 1)];

    int taskPerTh = (p_nGridSize + XBLOCK - 1) / XBLOCK;
    int idx = threadIdx.x;
    
    // First Assignment
    if (idx < p_nGridSize)
    {
        for (int i = 0; i < p_nEigNum - 1; i++)
        {
            MaxReduce[i*XBLOCK + idx] = p_devMax[idx + i * p_nGridSize];
            MinReduce[i*XBLOCK + idx] = p_devMin[idx + i * p_nGridSize];
        }
    }
    __syncthreads();

    // First Reduction
    for (int i = 1; i < taskPerTh; i++)
    {
        int curIndex = idx + i * XBLOCK;
        if (curIndex < p_nGridSize)
        {
            for (int j = 0; j < p_nEigNum - 1; j++)
            {
                float* maxAddr = &MaxReduce[j*XBLOCK + idx];
                float* minAddr = &MinReduce[j*XBLOCK + idx];
                atomicMax(maxAddr, p_devMax[curIndex + j * p_nGridSize]); // Use atomic operation for max
                atomicMin(minAddr, p_devMin[curIndex + j * p_nGridSize]); // Use atomic operation for min
            }
        }
    }
    __syncthreads();

    // Further Reductions
    int mask = 1;
    for (int level = 0; level < p_nMaxLevel; level++)
    {
        int index1 = idx;
        int index2 = (1 << level) + idx;
        if ((idx & mask) == 0 && index2 < p_nGridSize)
        {
            for (int i = 0; i < p_nEigNum - 1; i++)
            {
                float &maxRef = MaxReduce[i*XBLOCK + index1];
                float &minRef = MinReduce[i*XBLOCK + index1];
                atomicMax(&maxRef, MaxReduce[i*XBLOCK + index2]); // Use atomic operation for further reductions
                atomicMin(&minRef, MinReduce[i*XBLOCK + index2]); // Use atomic operation for further reductions
            }
        }
        __syncthreads();
        mask = (mask << 1) | 1;
    }

    // Write max and min into global memory
    if (idx == 0)
    {
        for (int i = 0; i < p_nEigNum - 1; i++)
        {
            p_devReduceMax[i] = MaxReduce[i*XBLOCK];
            p_devReduceMin[i] = MinReduce[i*XBLOCK];
        }
    }
}