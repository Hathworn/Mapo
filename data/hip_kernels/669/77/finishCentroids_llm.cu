#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void finishCentroids(int* centroidMass, unsigned int* centroidCount, float* centroids) {
    // Use variables to reduce redundant calculations
    int ty = blockIdx.y * blockDim.y + threadIdx.y;
    int tx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ty < 64 && tx < 34) {
        int idx = tx * 64 + ty;  // Pre-calculate index for memory access
        float mass = static_cast<float>(centroidMass[idx]);
        float totalCount = static_cast<float>(centroidCount[ty]);
        centroids[idx] = mass / (static_cast<float>(INTCONFACTOR) * totalCount);
    }
}