#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rgbUtoLab3F_kernel(int width, int height, float gamma, unsigned int* rgbU, float* devL, float* devA, float* devB) {
    int x0 = blockDim.x * blockIdx.x + threadIdx.x;
    int y0 = blockDim.y * blockIdx.y + threadIdx.y;

    if ((x0 < width) && (y0 < height)) {
        int index = y0 * width + x0;
        unsigned int rgb = rgbU[index];

        // Direct color channel calculation without intermediate variables
        float r = powf((float)(rgb & 0xff) / 255.0, gamma);
        float g = powf((float)((rgb & 0xff00) >> 8) / 255.0, gamma);
        float b = powf((float)((rgb & 0xff0000) >> 16) / 255.0, gamma);

        // Pre-computed coefficients multiplied directly with the components
        float x = (0.412453f * r) +  (0.357580f * g) + (0.180423f * b);
        float y = (0.212671f * r) +  (0.715160f * g) + (0.072169f * b);
        float z = (0.019334f * r) +  (0.119193f * g) + (0.950227f * b);

        // Constants moved outside loop as they don't change across threads
        const float x_ref = 0.950456f;
        const float y_ref = 1.000000f;
        const float z_ref = 1.088754f;
        const float threshold = 0.008856f;

        // Normalize color coordinates by reference values
        x /= x_ref;
        y /= y_ref;
        z /= z_ref;

        // Use ternary operator for inlining conditional logic
        float fx = (x > threshold) ? cbrtf(x) : (7.787f * x + (16.0f / 116.0f));
        float fy = (y > threshold) ? cbrtf(y) : (7.787f * y + (16.0f / 116.0f));
        float fz = (z > threshold) ? cbrtf(z) : (7.787f * z + (16.0f / 116.0f));

        // Calculate Lab components directly storing in global memory
        devL[index] = (y > threshold) ? (116 * cbrtf(y) - 16) : (903.3f * y);
        devA[index] = 500.0f * (fx - fy);
        devB[index] = 200.0f * (fy - fz);
    }
}