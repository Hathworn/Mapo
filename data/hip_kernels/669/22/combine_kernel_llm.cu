#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void combine_kernel(int nPixels, int cuePitchInFloats, float* devBg, float* devCga, float* devCgb, float* devTg, float* devMpb, float* devCombinedg) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int orientation = threadIdx.y;
    int orientedIndex = orientation * cuePitchInFloats + index;

    if (index < nPixels) {
        float accumulant = 0.0f;
        float accumulant2 = 0.0f;

        // Use loop unrolling for better performance
        #pragma unroll
        for (int i = 0; i < 3; ++i) {
            float* pointer = &devBg[orientedIndex + 8 * i * cuePitchInFloats];
            accumulant += *pointer * coefficients[i];
            accumulant2 += *pointer * weights[i];
        }

        #pragma unroll
        for (int i = 3; i < 6; ++i) {
            float* pointer = &devCga[orientedIndex + 8 * (i - 3) * cuePitchInFloats];
            accumulant += *pointer * coefficients[i];
            accumulant2 += *pointer * weights[i];
        }

        #pragma unroll
        for (int i = 6; i < 9; ++i) {
            float* pointer = &devCgb[orientedIndex + 8 * (i - 6) * cuePitchInFloats];
            accumulant += *pointer * coefficients[i];
            accumulant2 += *pointer * weights[i];
        }

        #pragma unroll
        for (int i = 9; i < 12; ++i) {
            float* pointer = &devTg[orientedIndex + 8 * (i - 9) * cuePitchInFloats];
            accumulant += *pointer * coefficients[i];
            accumulant2 += *pointer * weights[i];
        }

        devMpb[orientedIndex] = accumulant;
        devCombinedg[orientedIndex] = accumulant2;
    }
}