#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fillImage(int width, int height, int value, int* devOutput) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Precompute the index and check bounds only once
    if ((x < width) && (y < height)) {
        int index = y * width + x;
        devOutput[index] = value;
    }
}