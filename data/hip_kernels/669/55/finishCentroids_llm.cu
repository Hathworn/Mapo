#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void finishCentroids(int* centroidMass, unsigned int* centroidCount, float* centroids) {
    int centroidNumber = blockIdx.y * blockDim.y + threadIdx.y;
    int dimensionNumber = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to improve access efficiency
    __shared__ float sharedCentroidMass[32 * 34];
    __shared__ unsigned int sharedCentroidCount[32];

    if (centroidNumber < 32) {
        sharedCentroidCount[centroidNumber] = centroidCount[centroidNumber];
    }
    
    int index = dimensionNumber * 32 + centroidNumber;
    if (dimensionNumber < 34) {
        sharedCentroidMass[index] = (float)centroidMass[index];
    }

    __syncthreads(); // Ensure shared memory is fully populated before use

    if ((centroidNumber < 32) && (dimensionNumber < 34)) {
        float totalCount = (float)sharedCentroidCount[centroidNumber];
        float mass = sharedCentroidMass[index];
        centroids[index] = mass / ((float)INTCONFACTOR * totalCount);
    }
}