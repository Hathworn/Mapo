#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void finishCentroids_64(int* centroidMass, unsigned int* centroidCount, float* centroids) {
    // Calculate linear index for threads in grid
    int centroidNumber = blockIdx.y * blockDim.y + threadIdx.y;
    int dimensionNumber = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to minimize global memory access
    __shared__ unsigned int sharedCentroidCount[64];
    if (threadIdx.x == 0 && centroidNumber < 64) {
        sharedCentroidCount[centroidNumber] = centroidCount[centroidNumber];
    }
    __syncthreads();

    // Process only if within bounds
    if (centroidNumber < 64 && dimensionNumber < 34) {
        float totalCount = (float)sharedCentroidCount[centroidNumber];
        float mass = (float)centroidMass[dimensionNumber * 64 + centroidNumber];

        // Calculate and store result
        centroids[dimensionNumber * 64 + centroidNumber] = mass / ((float)INTCONFACTOR * totalCount);
    }
}