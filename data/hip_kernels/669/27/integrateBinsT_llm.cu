#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void integrateBinsT(int width, int height, int nbins, int binPitch, int* devIntegrals) {
    const int blockY = blockDim.y * blockIdx.x;
    const int threadY = threadIdx.y;
    const int bin = threadIdx.x;
    const int y = blockY + threadY;

    // Early exit to avoid out-of-bound processing
    if (y >= height || bin >= binPitch) return;

    // Calculate starting point once per thread
    int* imagePointer = devIntegrals + (binPitch * y + bin) * width;
    int accumulant = 0;

    // Prefetch next memory location to leverage memory coalescing
    for (int x = 0; x < width; x++) {
        accumulant += *imagePointer;
        *imagePointer = accumulant;
        imagePointer += binPitch;
    }
}