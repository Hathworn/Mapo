#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function optimized for better performance
__global__ void findDiffLabelsAtomicFree(float* devDiff, int diffPitchInFloats, int nPoints, int nClusters, int* devClusters, int* devChanges) {

    // Calculate global index
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if (x < nPoints) {
        int index = x;
        float minDistance = FLT_MAX;  // Use proper maximum for float
        int minCluster = -1;

        // Iterate through clusters to find the closest
        for(int cluster = 0; cluster < nClusters; cluster++) {
            float clusterDistance = devDiff[index];
            if (clusterDistance < minDistance) {
                minDistance = clusterDistance;
                minCluster = cluster;
            }
            index += diffPitchInFloats;
        }

        int previousCluster = devClusters[x];
        if (minCluster != previousCluster) {  // Update only if there's a change
            devClusters[x] = minCluster;
            atomicAdd(devChanges, 1);  // Safely increment changes using atomic
        }
    }
}