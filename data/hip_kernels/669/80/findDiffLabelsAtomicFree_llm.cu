#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findDiffLabelsAtomicFree(float* devDiff, int diffPitchInFloats, int nPoints, int nClusters, int* devClusters, int* devChanges) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if (x < nPoints) {
        int index = x;
        float minDistance = 10000000;
        int minCluster = -1;

        // Use register variables for loop invariants.
        for (int cluster = 0; cluster < nClusters; cluster++) {
            float clusterDistance = devDiff[index];
            if (clusterDistance < minDistance) {
                minDistance = clusterDistance;
                minCluster = cluster;
            }
            index += diffPitchInFloats;
        }

        int previousCluster = devClusters[x];
        devClusters[x] = minCluster;

        // Optimize the change check using local variable.
        if (minCluster != previousCluster) {
            atomicAdd(devChanges, 1);
        }
    }
}