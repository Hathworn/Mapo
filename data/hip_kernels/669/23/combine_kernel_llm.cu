#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void combine_kernel(int nPixels, int cuePitchInFloats, float* devBg, float* devCga, float* devCgb, float* devTg, float* devMpb, float* devCombinedg) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    int orientation = threadIdx.y;
    int orientedIndex = orientation * cuePitchInFloats + index;
    
    if (index < nPixels) {
        float accumulant = 0.0f;
        float accumulant2 = 0.0f;
        
        // Use a loop to reduce repetitive code
        float* inputs[4] = {devBg, devCga, devCgb, devTg};
        int coefIdx = 0;
        
        #pragma unroll
        for (int i = 0; i < 4; ++i) {
            float* basePointer = &inputs[i][orientedIndex];
            
            for (int j = 0; j < 3; ++j) {
                accumulant += basePointer[j * 8 * cuePitchInFloats] * coefficients[coefIdx];
                accumulant2 += basePointer[j * 8 * cuePitchInFloats] * weights[coefIdx];
                ++coefIdx;
            }
        }
        
        devMpb[orientedIndex] = accumulant;
        devCombinedg[orientedIndex] = accumulant2;
    }
}