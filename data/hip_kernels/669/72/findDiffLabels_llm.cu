#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findDiffLabels(float* devDiff, int diffPitchInFloats, int nPoints, int nClusters, int* devClusters, int* devChanges) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ unsigned int localChanges;

    // Initialize shared memory once per block for safety
    if (threadIdx.x == 0) localChanges = 0;
    __syncthreads();

    if (x < nPoints) {
        int index = x;
        float minDistance = 1.0e20f; // Use a large constant for initialization
        int minCluster = -1;

        // Use registers for minDistance and minCluster for faster access
        for (int cluster = 0; cluster < nClusters; cluster++) {
            float clusterDistance = devDiff[index];
            if (clusterDistance < minDistance) {
                minDistance = clusterDistance;
                minCluster = cluster;
            }
            index += diffPitchInFloats;
        }

        int previousCluster = devClusters[x];
        devClusters[x] = minCluster;

        // Use atomicAdd for consistent updates and limit atomic operations
        if (minCluster != previousCluster) {
            atomicAdd(&localChanges, 1);
        }
    }
    __syncthreads();

    // Combine localChanges to devChanges in a single atomicAdd
    if (threadIdx.x == 0) {
        atomicAdd(devChanges, localChanges);
    }
}