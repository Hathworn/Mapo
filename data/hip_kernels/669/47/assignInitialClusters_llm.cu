#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void assignInitialClusters(int width, int height, int nPixels, int clusterCount, int* cluster, int filterCount, float* responses, int* intResponses) {
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x >= width || y >= height) return;  // Exit early if out of bounds

    int pixel = y * width + x;

    // Precompute block sizes and limit assigned cluster
    int xBlock = x / ((width - 1) / 6 + 1);
    int yBlock = y / ((height - 1) / 6 + 1);
    int assignedCluster = min(yBlock * 6 + xBlock, 31);

    // Assign cluster
    cluster[pixel] = assignedCluster;

    // Optimize loop with SIMD-like pattern
    for (int i = 0; i < filterCount; i++) {
        int index = pixel + i * nPixels;
        intResponses[index] = __float2int_rz(INTCONFACTOR * responses[index]);
    }
}