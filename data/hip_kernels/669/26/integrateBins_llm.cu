#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void integrateBins(int width, int height, int nbins, int* devImage, int binPitch, int* devIntegrals) {
    __shared__ int pixels[16];
    const int blockX = blockDim.y * blockIdx.x;
    const int threadX = threadIdx.y;
    const int bin = threadIdx.x;
    const int x = blockX + threadX;
    
    // Exit if block-level `x` coordinate exceeds image width
    if (x >= width || bin >= nbins) return;
    
    int* imagePointer = devImage + x;
    int* outputPointer = devIntegrals + binPitch * x + bin;
    int accumulant = 0;
    
    // Iterate over image rows
    for (int y = 0; y < height; y++) {
        // Each block initializes shared memory for its portion
        if (bin == 0) {
            pixels[threadX] = *imagePointer;
        }
        
        // Wait for all threads to load current pixel row
        __syncthreads();
        
        // Check for bin match and accumulate
        if (pixels[threadX] == bin) {
            accumulant++;
        }
        
        // Store accumulator value in output
        *outputPointer = accumulant;
        
        // Move to next row in memory
        imagePointer += width;
        outputPointer += width * binPitch;
    }
}