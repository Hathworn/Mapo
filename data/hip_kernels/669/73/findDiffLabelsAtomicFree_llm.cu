#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findDiffLabelsAtomicFree(float* devDiff, int diffPitchInFloats, int nPoints, int nClusters, int* devClusters, int* devChanges) {
    // Calculate global thread index
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    // Check if thread index is within range of points
    if (x < nPoints) {
        int index = x;
        float minDistance = FLT_MAX; // Use constant for clarity
        int minCluster = -1;

        // Unroll loop to improve performance if nClusters is small and known
        for (int cluster = 0; cluster < nClusters; ++cluster) {
            float clusterDistance = devDiff[index];
            if (clusterDistance < minDistance) {
                minDistance = clusterDistance;
                minCluster = cluster;
            }
            index += diffPitchInFloats;
        }

        // Update clusters and calculate changes
        int previousCluster = devClusters[x];
        devClusters[x] = minCluster;
        if (minCluster != previousCluster) {
            atomicAdd(devChanges, 1); // Use atomicAdd for thread safety
        }
    }
}