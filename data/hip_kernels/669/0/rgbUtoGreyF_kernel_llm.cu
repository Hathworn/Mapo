#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function for better performance
__global__ void rgbUtoGreyF_kernel(int width, int height, unsigned int* rgbU, float* grey) {
    // Calculate the column and row index for the current thread
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    int y = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure the thread is within image bounds
    if ((x < width) && (y < height)) {
        // Compute the index for 1D array access
        int index = y * width + x;

        // Retrieve RGB value and extract individual channels
        unsigned int rgb = rgbU[index];
        float r = (rgb & 0xff) * 0.29894f / 255.0f;
        float g = ((rgb & 0xff00) >> 8) * 0.58704f / 255.0f;
        float b = ((rgb & 0xff0000) >> 16) * 0.11402f / 255.0f;

        // Calcualte grayscale value using weighted sum
        grey[index] = r + g + b;
    }
}