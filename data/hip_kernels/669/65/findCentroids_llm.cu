#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findCentroids(int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount) {
    __shared__ int localMasses[64 * 17];
    __shared__ unsigned int localCounts[64];
    int pixel = blockDim.x * blockIdx.x + threadIdx.x;

    // Initialize shared memory in parallel
    for (int i = threadIdx.x; i < 64 * 17; i += blockDim.x) {
        localMasses[i] = 0;
    }
    for (int i = threadIdx.x; i < 64; i += blockDim.x) {
        localCounts[i] = 0;
    }
    __syncthreads();

    if (pixel < nPixels) {
        int myCluster = cluster[pixel];
        int myIndex = pixel;
        
        // Process first set of filters
        for (int filter = 0; filter < 17; filter++) {
            int myElement = responses[myIndex];
            atomicAdd(localMasses + filter * 64 + myCluster, myElement);
            myIndex += nPixels;
        }
    }
    __syncthreads();

    // Reduce localMasses and reset them
    for (int i = threadIdx.x; i < 64 * 17; i += blockDim.x) {
        atomicAdd(centroidMass + i, localMasses[i]);
        localMasses[i] = 0;
    }
    __syncthreads();

    if (pixel < nPixels) {
        int myCluster = cluster[pixel];
        int myIndex = pixel + nPixels * 17;

        // Process second set of filters
        for (int filter = 17; filter < 34; filter++) {
            int myElement = responses[myIndex];
            atomicAdd(localMasses + filter * 64 + myCluster, myElement);
            myIndex += nPixels;
        }
        atomicInc(localCounts + myCluster, 100000000);
    }
    __syncthreads();

    // Reduce localMasses and localCounts
    for (int i = threadIdx.x; i < 64 * 17; i += blockDim.x) {
        if (i < 64 * (34 - 17)) {
            atomicAdd(centroidMass + i + 64 * 17, localMasses[i + 64 * 17]);
        }
    }

    for (int i = threadIdx.x; i < 64; i += blockDim.x) {
        atomicAdd(centroidCount + i, localCounts[i]);
    }
}