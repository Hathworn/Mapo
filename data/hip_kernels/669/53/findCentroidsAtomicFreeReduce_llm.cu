#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findCentroidsAtomicFreeReduce(int afLocal, int* responses, int nPixels, int* cluster, int* centroidMass, unsigned int* centroidCount)
{
    // Identify unique thread and block indices
    int const af_id = blockIdx.x;
    int const cluster_id = blockIdx.y;
    int const filter_id = threadIdx.x;

    // Initialize shared memory for mass and count
    __shared__ int sharedMass[32];
    __shared__ unsigned int sharedCount[32];

    // Initialize local variables
    int local_mass = 0;
    unsigned int local_count = 0;

    // Load initial values to shared memory
    if (af_id == 0)
    {
        int idx0 = filter_id * 32 + cluster_id;
        
        for (int i = 0; i < gridDim.x; i++)
        {
            int idxother = i * gridDim.y * blockDim.x + idx0;
            local_mass += centroidMass[idxother];
            local_count += centroidCount[idxother];
        }

        // Store computed values in shared memory
        sharedMass[filter_id] = local_mass;
        sharedCount[filter_id] = local_count;

        // Synchronize threads
        __syncthreads();
        
        // Write back the results from shared memory
        centroidMass[idx0] = sharedMass[filter_id];
        centroidCount[idx0] = sharedCount[filter_id];
    }
}