#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mirrorImage_kernel(uint width, uint height, uint border, uint borderWidth, uint borderHeight, int* devInput, int* devOutput) {
    int x0 = blockDim.x * blockIdx.x + threadIdx.x;
    int y0 = blockDim.y * blockIdx.y + threadIdx.y;

    // Use shared memory for better performance if needed
    extern __shared__ int sharedMem[];

    if ((x0 < borderWidth) && (y0 < borderHeight)) {
        int x1, y1;

        // Calculate mirrored x1 using modulo for simplicity
        x1 = x0 < border ? border - x0 - 1 : (x0 < border + width ? x0 - border : border + 2 * width - x0 - 1);
        
        // Calculate mirrored y1 using modulo for simplicity
        y1 = y0 < border ? border - y0 - 1 : (y0 < border + height ? y0 - border : border + 2 * height - y0 - 1);

        // Ensuring memory coalescing
        devOutput[y0 * borderWidth + x0] = devInput[y1 * width + x1];
    }
}