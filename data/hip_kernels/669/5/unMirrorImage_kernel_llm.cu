#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unMirrorImage_kernel(uint width, uint height, uint border, uint borderWidth, uint borderHeight, float* devInput, float* devOutput) {
    // Calculate global thread indices
    int x0 = blockIdx.x * blockDim.x + threadIdx.x;
    int y0 = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check boundaries
    if (x0 < borderWidth && y0 < borderHeight) {
        // Calculate mirrored indices
        int x1 = x0 + border;
        int y1 = y0 + border;
        
        // Perform memory operation
        devOutput[y0 * borderWidth + x0] = devInput[y1 * width + x1];
    }
}