#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex)
{
    __shared__ int uni;

    // Load uniforms value into shared memory once per block
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];
    __syncthreads();  // Ensure all threads have the updated value

    // Calculate the address with efficient multiplication
    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x;

    // Perform two adds per thread only if within bounds
    if (address < n) {
        g_data[address] += uni;
        if (address + blockDim.x < n)
            g_data[address + blockDim.x] += uni;
    }
}