#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeBoundary_kernel(int startPos, int rLen, int* d_startArray, int* d_startSumArray, int* d_bounary)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = startPos + tid;

    // Ensure thread is within bounds
    if (pos < rLen) {
        int flag = d_startArray[pos];
        int writePos = d_startSumArray[pos];
        // Write boundary conditionally
        if (flag == 1) {
            d_bounary[writePos] = pos;
        }
    }
}