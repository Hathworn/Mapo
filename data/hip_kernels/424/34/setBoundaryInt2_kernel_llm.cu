#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setBoundaryInt2_kernel(int* d_boundary, int startPos, int numKey, int rLen, int2* d_boundaryRange)
{
    // Calculate global thread ID
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate global position
    int pos = startPos + tid;
    
    // Check if within bounds
    if(pos < numKey)
    {
        int2 flag;
        flag.x = d_boundary[pos];
        
        // Avoid boundary check
        flag.y = (pos + 1 != numKey) ? d_boundary[pos + 1] : rLen;
        
        // Assign to output
        d_boundaryRange[pos] = flag;
    }
}