#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "setBoundaryInt2_kernel_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    int *d_boundary = NULL;
hipMalloc(&d_boundary, XSIZE*YSIZE);
int startPos = 1;
int numKey = 1;
int rLen = 1;
int2 *d_boundaryRange = NULL;
hipMalloc(&d_boundaryRange, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
setBoundaryInt2_kernel<<<gridBlock, threadBlock>>>(d_boundary,startPos,numKey,rLen,d_boundaryRange);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
setBoundaryInt2_kernel<<<gridBlock, threadBlock>>>(d_boundary,startPos,numKey,rLen,d_boundaryRange);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
setBoundaryInt2_kernel<<<gridBlock, threadBlock>>>(d_boundary,startPos,numKey,rLen,d_boundaryRange);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}