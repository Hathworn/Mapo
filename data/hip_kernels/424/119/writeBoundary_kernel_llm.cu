#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeBoundary_kernel(int startPos, int rLen, int* d_startArray, int* d_startSumArray, int* d_bounary)
{
    // Calculate the global thread index directly
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int pos = startPos + tid;
    
    // Ensure thread index does not exceed the range
    if (pos < rLen) {
        int flag = d_startArray[pos];
        int writePos = d_startSumArray[pos];
        
        // Write boundary if the flag condition is met
        if (flag == 1) {
            d_bounary[writePos] = pos;
        }
    }
}