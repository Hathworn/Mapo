#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setBoundaryInt2_kernel(int* d_boundary, int startPos, int numKey, int rLen, int2* d_boundaryRange)
{
    // Use one-dimensional indexing for blocks
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate global position
    int pos = startPos + tid;

    if (pos < numKey)
    {
        int2 flag;
        flag.x = d_boundary[pos];
        // Avoid branching by using conditional operator
        flag.y = (pos + 1 < numKey) ? d_boundary[pos + 1] : rLen;
        d_boundaryRange[pos] = flag;
    }
}