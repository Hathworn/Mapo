#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getIntYArray_kernel(int2* d_input, int startPos, int rLen, int* d_output)
{
    // Calculate unique thread ID
    int tid = blockIdx.x * blockDim.x * gridDim.y + blockIdx.y * blockDim.x + threadIdx.x + threadIdx.y * blockDim.x;
    int pos = startPos + tid;

    // Only proceed if within range
    if (pos < rLen)
    {
        d_output[pos] = d_input[pos].y; // Direct access to .y component
    }
}