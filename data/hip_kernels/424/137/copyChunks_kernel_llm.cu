#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyChunks_kernel(void *d_source, int startPos, int2* d_Rin, int rLen, int *d_sum, void *d_dest)
{
    // Index calculations for block and thread
    const int bx = blockIdx.x;
    const int tx = threadIdx.x;
    const int tid = tx + blockIdx.y * blockDim.x;
    const int resultID = bx * gridDim.y * blockDim.x + tid;
    int pos = startPos + resultID;

    // Check bounds before proceeding
    if (pos < rLen)
    {
        int2 value = d_Rin[pos];
        int offset = value.x;
        int size = value.y;
        int startWritePos = d_sum[pos];
        
        // Use pointer arithmetic for memory copy
        char *source = (char*)d_source + offset;
        char *dest = (char*)d_dest + startWritePos;
        
        // Efficient memory copy
        for (int i = 0; i < size; i++)
        {
            dest[i] = source[i];
        }
        
        // Update start position
        value.x = startWritePos;
        d_Rin[pos] = value;
    }
}