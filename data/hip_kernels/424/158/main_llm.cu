#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "copyChunks_kernel_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    void *d_source = NULL;
hipMalloc(&d_source, XSIZE*YSIZE);
int startPos = 1;
int2 *d_Rin = NULL;
hipMalloc(&d_Rin, XSIZE*YSIZE);
int rLen = 1;
int *d_sum = NULL;
hipMalloc(&d_sum, XSIZE*YSIZE);
void *d_dest = NULL;
hipMalloc(&d_dest, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
copyChunks_kernel<<<gridBlock, threadBlock>>>(d_source,startPos,d_Rin,rLen,d_sum,d_dest);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
copyChunks_kernel<<<gridBlock, threadBlock>>>(d_source,startPos,d_Rin,rLen,d_sum,d_dest);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
copyChunks_kernel<<<gridBlock, threadBlock>>>(d_source,startPos,d_Rin,rLen,d_sum,d_dest);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}