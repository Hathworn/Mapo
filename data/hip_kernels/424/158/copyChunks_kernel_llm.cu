#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyChunks_kernel(void *d_source, int startPos, int2* d_Rin, int rLen, int *d_sum, void *d_dest)
{
    const int tid = threadIdx.y * blockDim.x + threadIdx.x;  // Flatten thread index
    const int bid = blockIdx.y * gridDim.x + blockIdx.x;     // Flatten block index
    const int numThread = blockDim.x * blockDim.y;           // Calculate total threads in block
    const int resultID = bid * numThread + tid;              // Calculate global index
    int pos = startPos + resultID;

    if (pos < rLen)
    {
        int2 value = d_Rin[pos];
        int offset = value.x;
        int size = value.y;
        int startWritePos = d_sum[pos];
        char *source = static_cast<char*>(d_source);
        char *dest = static_cast<char*>(d_dest);

        // Optimize memory copy using memcpy
        memcpy(dest + startWritePos, source + offset, size * sizeof(char));

        value.x = startWritePos;
        d_Rin[pos] = value;
    }
}