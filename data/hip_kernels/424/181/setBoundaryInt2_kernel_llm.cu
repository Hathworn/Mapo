#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setBoundaryInt2_kernel(int* d_boundary, int startPos, int numKey, int rLen, int2* d_boundaryRange)
{
    // Calculate global thread index
    int resultID = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = startPos + resultID;

    // Ensure the position is within bounds
    if (pos < numKey)
    {
        int2 flag;
        flag.x = d_boundary[pos];
        flag.y = (pos + 1 != numKey) ? d_boundary[pos + 1] : rLen; // Set boundary or rLen
        d_boundaryRange[pos] = flag;
    }
}