#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getIntYArray_kernel(int2* d_input, int startPos, int rLen, int* d_output)
{
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x + threadIdx.y * gridDim.x * blockDim.x * blockDim.y;

    int pos = startPos + tid;
    if (pos < rLen)
    {
        int2 value = d_input[pos];
        d_output[pos] = value.y;
    }
}