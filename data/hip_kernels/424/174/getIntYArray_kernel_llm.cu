#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getIntYArray_kernel(int2* d_input, int startPos, int rLen, int* d_output)
{
    // Compute a global thread index
    int idx = blockIdx.x * blockDim.x * gridDim.y + blockIdx.y * blockDim.x + threadIdx.x + threadIdx.y * blockDim.x;

    // Calculate position from startPos and index
    int pos = startPos + idx;

    // Ensure position is within range and assign output
    if (pos < rLen)
    {
        d_output[pos] = d_input[pos].y; // Directly access the 'y' field
    }
}