#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex)
{
    __shared__ int uni;
    // Load the uniform into shared memory only once per block
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];

    __syncthreads(); // Ensure the uniform value is loaded before use

    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x;

    // Use logical AND to prevent out-of-bound access and reduce branch divergence
    if (threadIdx.x + blockDim.x < n) {
        g_data[address]              += uni;
        g_data[address + blockDim.x] += uni;
    } else {
        g_data[address] += uni;
    }
}