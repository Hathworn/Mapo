#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getIntYArray_kernel(int2* d_input, int startPos, int rLen, int* d_output)
{
    // Compute unique global thread ID
    int gid = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x + threadIdx.y * gridDim.x * blockDim.y;
    int pos = startPos + gid;

    // Ensure position is within bounds before writing to the output
    if (pos < rLen)
    {
        d_output[pos] = d_input[pos].y;
    }
}