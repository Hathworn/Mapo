#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setBoundaryInt2_kernel(int* d_boundary, int startPos, int numKey, int rLen, int2* d_boundaryRange)
{
    // Use blockDim.x * blockDim.y for linear thread ID calculation
    const int bid = blockIdx.x + blockIdx.y * gridDim.x;
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;
    const int numThread = blockDim.x * blockDim.y; // Total threads per block
    const int resultID = bid * numThread + tid;
    int pos = startPos + resultID;

    if(pos < numKey)
    {
        int2 flag;
        flag.x = d_boundary[pos];
        flag.y = (pos + 1 != numKey) ? d_boundary[pos + 1] : rLen; // Use ternary for boundary check
        d_boundaryRange[pos] = flag;
    }
}