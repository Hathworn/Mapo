#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setBoundaryInt2_kernel(int* d_boundary, int startPos, int numKey, int rLen, int2* d_boundaryRange)
{
    // Use blockDim.x * blockDim.y for more accurate numThreads calculation
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;
    const int bid = blockIdx.x + blockIdx.y * gridDim.x;
    const int numThreads = blockDim.x * blockDim.y;
    const int resultID = bid * numThreads + tid;
    int pos = startPos + resultID;

    if (pos < numKey)
    {
        // Simplify condition inside loop
        int2 flag;
        flag.x = d_boundary[pos];
        flag.y = (pos + 1 != numKey) ? d_boundary[pos + 1] : rLen;
        d_boundaryRange[pos] = flag;
    }
}