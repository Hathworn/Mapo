#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeBoundary_kernel(int startPos, int rLen, int* d_startArray, int* d_startSumArray, int* d_bounary)
{
    // Use 1D grid and block for better performance
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = startPos + tid;

    if (pos < rLen)
    {
        int flag = d_startArray[pos];
        int writePos = d_startSumArray[pos];
        if (flag == 1)
            d_bounary[writePos] = pos;
    }
}