#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex)
{
    __shared__ int uni;
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];

    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x;

    __syncthreads();

    // Efficient vectorized addition using a single add per thread
    if (address < n)
        g_data[address] += uni;
    if (address + blockDim.x < n)
        g_data[address + blockDim.x] += uni;
}