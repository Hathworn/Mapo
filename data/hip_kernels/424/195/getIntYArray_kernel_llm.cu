#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getIntYArray_kernel(int2* d_input, int startPos, int rLen, int* d_output)
{
    // Calculate global thread ID
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Calculate position
    int pos = startPos + tid;
    
    // Ensure position is within bounds
    if (pos < rLen)
    {
        int2 value = d_input[pos];
        d_output[pos] = value.y;
    }
}