#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setBoundaryInt2_kernel(int* d_boundary, int startPos, int numKey, int rLen, int2* d_boundaryRange) {
    // Assign unique thread identifier
    int pos = startPos + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y) + threadIdx.x + threadIdx.y * blockDim.x;

    // Check if the position is within the range
    if (pos < numKey) {
        // Set values for boundary range
        d_boundaryRange[pos] = make_int2(d_boundary[pos], (pos + 1 < numKey) ? d_boundary[pos + 1] : rLen);
    }
}