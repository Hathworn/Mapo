#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeBoundary_kernel(int startPos, int rLen, int* d_startArray, int* d_startSumArray, int* d_bounary)
{
    // Optimize thread index calculation using 1D block and grid indexing
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = startPos + index;
    
    if (pos < rLen) {
        int flag = d_startArray[pos];
        int writePos = d_startSumArray[pos];
        if (flag == 1)
            d_bounary[writePos] = pos;
    }
}