#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setBoundaryInt2_kernel(int* d_boundary, int startPos, int numKey, int rLen, int2* d_boundaryRange)
{
    // Optimize thread index calculation
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;
    const int bid = blockIdx.x + blockIdx.y * gridDim.x;
    const int numThread = blockDim.x * blockDim.y;  // Use 2D block size

    const int resultID = bid * numThread + tid;
    int pos = startPos + resultID;

    if (pos < numKey)
    {
        int2 flag;
        flag.x = d_boundary[pos];
        flag.y = (pos + 1 < numKey) ? d_boundary[pos + 1] : rLen;  // Simplify conditional operation
        d_boundaryRange[pos] = flag;
    }
}