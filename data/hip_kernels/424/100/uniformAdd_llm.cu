#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex)
{
    __shared__ int uni;
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];

    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x;

    __syncthreads();

    // Simplified addressing and removed unnecessary conditional
    g_data[address] += uni;
    if (threadIdx.x + blockDim.x < n) // Avoid unnecessary multiplication
        g_data[address + blockDim.x] += uni;
}