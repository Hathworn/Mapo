#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getIntYArray_kernel(int2* d_input, int startPos, int rLen, int* d_output)
{
    // Calculate unique thread index
    int pos = startPos + blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds
    if (pos < rLen)
    {
        // Directly access d_input and assign to d_output
        d_output[pos] = d_input[pos].y;
    }
}