#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeBoundary_kernel(int startPos, int rLen, int* d_startArray, int* d_startSumArray, int* d_bounary)
{
    // Calculate unique global thread index
    int resultID = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = startPos + resultID;

    // Check if within bounds
    if (pos < rLen)
    {
        int flag = d_startArray[pos]; // Load flag
        int writePos = d_startSumArray[pos]; // Load write position
        if (flag == 1) {
            d_bounary[writePos] = pos; // Write position if flag is 1
        }
    }
}