#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyChunks_kernel(void *d_source, int startPos, int2* d_Rin, int rLen, int *d_sum, void *d_dest)
{
    const int bid = blockIdx.y * gridDim.x + blockIdx.x; // Improved block index calculation
    const int tid = threadIdx.y * blockDim.x + threadIdx.x; // Improved thread index calculation
    const int resultID = bid * (blockDim.x * blockDim.y) + tid; // Ensure correct global thread ID
    int pos = startPos + resultID;

    if (pos < rLen)
    {
        int2 value = d_Rin[pos];
        int offset = value.x;
        int size = value.y;
        int startWritePos = d_sum[pos];
        char *source = reinterpret_cast<char*>(d_source); // Use reinterpret_cast for clarity
        char *dest = reinterpret_cast<char*>(d_dest); // Use reinterpret_cast for clarity

        // Utilize pointer arithmetic instead of loop index
        char *sourcePtr = source + offset;
        char *destPtr = dest + startWritePos;

        // Use a loop unrolling technique for potential performance gain
        #pragma unroll
        for (int i = 0; i < size; ++i)
        {
            destPtr[i] = sourcePtr[i];
        }
        
        value.x = startWritePos;
        d_Rin[pos] = value;
    }
}