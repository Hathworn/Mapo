#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyChunks_kernel(void *d_source, int startPos, int2* d_Rin, int rLen, int *d_sum, void *d_dest)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread ID directly

    if (tid < rLen)  // Only execute if within bounds
    {
        int2 value = d_Rin[tid];
        int offset = value.x;
        int size = value.y;
        int startWritePos = d_sum[tid];
        char *source = (char*)d_source + offset;  // Pointer arithmetic to avoid recalculating inside loop
        char *dest = (char*)d_dest + startWritePos;

        for (int i = 0; i < size; i++)  // Simplify the loop 
        {
            dest[i] = source[i];
        }
        
        d_Rin[tid].x = startWritePos;  // Store updated start position
    }
}