#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyChunks_kernel(void *d_source, int startPos, int2* d_Rin, int rLen, int *d_sum, void *d_dest)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x; // Simplified thread index calculation

    if(tid < rLen) // Use single thread index for bounds check
    {
        int2 value = d_Rin[tid];  // Access only the required index
        int offset = value.x;
        int size = value.y;
        int startWritePos = d_sum[tid]; 
        char *source = (char*)d_source + offset; // Pre-calculate source pointer
        char *dest = (char*)d_dest + startWritePos; // Pre-calculate destination pointer

        for(int i = 0; i < size; ++i)
        {
            dest[i] = source[i]; // Use pre-calculated pointers in copy operation
        }

        value.x = startWritePos;
        d_Rin[tid] = value; // Write back only the modified value
    }
}