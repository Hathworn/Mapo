#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeBoundary_kernel(int startPos, int rLen, int* d_startArray, int* d_startSumArray, int* d_bounary)
{
    // Use a single dimension for grid and block, simplifying index calculations
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = startPos + tid;

    // Ensure only threads with valid positions execute logic
    if (pos < rLen)
    {
        int flag = d_startArray[pos];
        int writePos = d_startSumArray[pos];
        if (flag == 1)
        {
            d_bounary[writePos] = pos;
        }
    }
}