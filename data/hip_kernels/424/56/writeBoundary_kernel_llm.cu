#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeBoundary_kernel(int startPos, int rLen, int* d_startArray, int* d_startSumArray, int* d_bounary)
{
    // Calculate the global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int pos = startPos + tid;

    if(pos < rLen) // Use single dimension indexing for better performance
    {
        int flag = d_startArray[pos];
        int writePos = d_startSumArray[pos];
        if(flag == 1)
        {
            d_bounary[writePos] = pos;
        }
    }
}