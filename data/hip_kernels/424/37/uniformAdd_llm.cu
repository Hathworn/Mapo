#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex)
{
    __shared__ int uni;
    
    // Load uniforms to shared memory only once per block
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];

    // Calculate global thread address
    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x;

    __syncthreads();

    // Update data with uniform value; check address bounds for second add
    g_data[address] += uni;
    if (threadIdx.x + blockDim.x < n)
        g_data[address + blockDim.x] += uni;
}