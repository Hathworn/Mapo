#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getIntYArray_kernel(int2* d_input, int startPos, int rLen, int* d_output)
{
    // Use a 1D block and grid to simplify the index calculation
    int pos = startPos + blockIdx.x * blockDim.x + threadIdx.x;
    
    if(pos < rLen)
    {
        // Fetch and store y component of int2
        d_output[pos] = d_input[pos].y;
    }
}