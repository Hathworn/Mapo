#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyChunks_kernel(void *d_source, int startPos, int2* d_Rin, int rLen, int *d_sum, void *d_dest)
{
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;
    const int bid = blockIdx.x + blockIdx.y * gridDim.x;
    const int numThread = blockDim.x * blockDim.y; // Using both blockDim.x and blockDim.y
    const int resultID = bid * numThread + tid; // Calculating global thread index
    int pos = startPos + resultID;

    if (pos < rLen)
    {
        int2 value = d_Rin[pos];
        int offset = value.x;
        int size = value.y;
        int startWritePos = d_sum[pos];
        char* source = (char*)d_source + offset; // Adjust pointer directly
        char* dest = (char*)d_dest + startWritePos; // Adjust pointer directly

        for (int i = 0; i < size; i++)
        {
            dest[i] = source[i]; // Simplified copying logic
        }
        value.x = startWritePos;
        d_Rin[pos] = value; // Write back the updated start write position
    }
}