#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setBoundaryInt2_kernel(int* d_boundary, int startPos, int numKey, int rLen, int2* d_boundaryRange)
{
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;
    const int bid = blockIdx.x + blockIdx.y * gridDim.x;

    // Calculate global thread index and position
    const int resultID = bid * blockDim.x + tid;
    int pos = startPos + resultID;

    // Update boundary range only if within bounds
    if (pos < numKey)
    {
        int2 flag;
        flag.x = d_boundary[pos];
        flag.y = (pos + 1 != numKey) ? d_boundary[pos + 1] : rLen;
        d_boundaryRange[pos] = flag;
    }
}