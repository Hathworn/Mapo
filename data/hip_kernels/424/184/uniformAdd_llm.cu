#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex)
{
    __shared__ int uni;
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];

    // Better indexing with simpler operations
    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x;

    __syncthreads();
    
    // Use conditional with a unified index calculation
    if (address < n) {
        g_data[address] += uni;
    }
    if (address + blockDim.x < n) {
        g_data[address + blockDim.x] += uni;
    }
}