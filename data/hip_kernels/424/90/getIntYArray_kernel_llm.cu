#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getIntYArray_kernel(int2* d_input, int startPos, int rLen, int* d_output)
{
    const int tid = blockIdx.x * blockDim.x + threadIdx.x; // Compute global thread index directly
    const int pos = startPos + tid; // Calculate position directly with computed index
    if(pos < rLen) // Guard the operation to avoid out-of-bounds access
    {
        int2 value = d_input[pos];
        d_output[pos] = value.y; // Directly store the result 
    }
}