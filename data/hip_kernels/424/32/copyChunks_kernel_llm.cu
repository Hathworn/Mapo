#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyChunks_kernel(void *d_source, int startPos, int2* d_Rin, int rLen, int *d_sum, void *d_dest)
{
    // Calculate global thread ID using block and thread indices
    int globalThreadId = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    int pos = startPos + globalThreadId;

    // Check if thread should perform operation, avoiding out-of-bounds access
    if(pos < rLen)
    {
        // Load value from d_Rin and perform copy operation
        int2 value = d_Rin[pos];
        int offset = value.x;
        int size = value.y;
        int startWritePos = d_sum[pos];

        // Use shared memory for improved performance in copying
        extern __shared__ char sharedSource[];
        char *source = (char*)d_source;
        char *dest = (char*)d_dest;

        // Fill shared memory with chunk data
        for(int i = 0; i < size; i++)
        {
            sharedSource[i] = source[i + offset];
        }
        __syncthreads(); // Synchronize threads before writing to global memory

        // Write back to the global memory from shared memory
        for(int i = 0; i < size; i++)
        {
            dest[i + startWritePos] = sharedSource[i];
        }

        // Update the position for d_Rin
        value.x = startWritePos;
        d_Rin[pos] = value;
    }
}