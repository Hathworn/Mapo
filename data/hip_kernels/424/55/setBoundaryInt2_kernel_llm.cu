#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setBoundaryInt2_kernel(int* d_boundary, int startPos, int numKey, int rLen, int2* d_boundaryRange)
{
    // Calculate thread and block indices using built-in block and thread dimensions
    const int tid = threadIdx.x + blockDim.x * blockIdx.x + blockDim.x * blockIdx.y * gridDim.x;
    const int pos = startPos + tid;

    // Ensure the position is within the number of keys
    if (pos < numKey)
    {
        int2 flag;
        flag.x = d_boundary[pos];
        // Check if pos is not the last, then use next element, otherwise use rLen
        flag.y = (pos + 1 != numKey) ? d_boundary[pos + 1] : rLen;
        d_boundaryRange[pos] = flag;
    }
}