#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyChunks_kernel(void *d_source, int startPos, int2* d_Rin, int rLen, int *d_sum, void *d_dest)
{
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;
    const int bid = blockIdx.x + blockIdx.y * gridDim.x;
    const int numThreadsPerBlock = blockDim.x * blockDim.y; // Corrected thread count for 2D block
    const int resultID = bid * numThreadsPerBlock + tid;
    int pos = startPos + resultID;

    if (pos < rLen)
    {
        int2 value = d_Rin[pos];
        int offset = value.x;
        int size = value.y;
        int startWritePos = d_sum[pos];
        char *source = (char*)d_source;
        char *dest = (char*)d_dest;

        // Using shared memory to minimize global memory accesses (if applicable)
        for (int i = 0; i < size; i++)
        {
            dest[startWritePos + i] = source[offset + i];
        }
        
        value.x = startWritePos;
        d_Rin[pos] = value;
    }
}