#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeBoundary_kernel(int startPos, int rLen, int* d_startArray, int* d_startSumArray, int* d_bounary)
{
    // Calculate thread and block indices more efficiently
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;
    const int bid = blockIdx.x + blockIdx.y * gridDim.x;
    const int resultID = bid * blockDim.x * blockDim.y + tid; // Combine blockDim.x and blockDim.y
    int pos = startPos + resultID;

    // Process only threads that have valid positions
    if (pos < rLen)
    {
        int flag = d_startArray[pos];
        int writePos = d_startSumArray[pos];
        
        // Conditional store only if flag equals 1
        if (flag == 1)
        {
            d_bounary[writePos] = pos;
        }
    }
}