#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeBoundary_kernel(int startPos, int rLen, int* d_startArray, int* d_startSumArray, int* d_bounary)
{
    // Calculate global thread ID
    int tid = threadIdx.x + blockIdx.x * blockDim.x + (threadIdx.y + blockIdx.y * blockDim.y) * gridDim.x * blockDim.x;
    int pos = startPos + tid;

    // Ensure pos is within range
    if (pos < rLen)
    {
        int flag = d_startArray[pos];
        int writePos = d_startSumArray[pos];
        if (flag == 1)
            d_bounary[writePos] = pos;
    }
}