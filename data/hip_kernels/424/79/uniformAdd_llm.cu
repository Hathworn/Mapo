#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) 
{
    __shared__ int uni;
    
    // Load uniforms value into shared memory
    if (threadIdx.x == 0) {
        uni = uniforms[blockIdx.x + blockOffset];
    }

    // Calculate global memory address
    unsigned int address = blockIdx.x * blockDim.x * 2 + baseIndex + threadIdx.x;

    __syncthreads();

    // Perform uniform add
    if (address < n) {
        g_data[address] += uni;
        if (address + blockDim.x < n) {
            g_data[address + blockDim.x] += uni;
        }
    }
}