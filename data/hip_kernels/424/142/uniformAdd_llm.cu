#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex)
{
    __shared__ int uni;

    // Load the uniform value only once into shared memory by the first thread
    if (threadIdx.x == 0) {
        uni = uniforms[blockIdx.x + blockOffset];
    }
    
    __syncthreads(); // Ensure uniform value is loaded before proceeding

    // Optimize address calculation using intrinsic function
    unsigned int address = __fma_rn(blockIdx.x, (blockDim.x << 1), baseIndex + threadIdx.x);

    // Update g_data array with the value in shared memory
    g_data[address] += uni;
    if (threadIdx.x + blockDim.x < n) {
        g_data[address + blockDim.x] += uni;
    }
}