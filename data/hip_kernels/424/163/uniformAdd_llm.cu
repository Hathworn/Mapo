#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex)
{
    __shared__ int uni;
    
    // Load uniform value to shared memory for the block
    if (threadIdx.x == 0) 
        uni = uniforms[blockIdx.x + blockOffset];
    
    // Calculate the global memory address for the thread
    unsigned int address = blockIdx.x * (blockDim.x * 2) + baseIndex + threadIdx.x; 
    
    __syncthreads();
    
    // Add uniform to both positions, check boundary for the second add
    if(address < n)
        g_data[address] += uni;
    if(address + blockDim.x < n)
        g_data[address + blockDim.x] += uni;
}