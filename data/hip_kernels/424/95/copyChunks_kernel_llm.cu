#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyChunks_kernel(void *d_source, int startPos, int2* d_Rin, int rLen, int *d_sum, void *d_dest)
{
    const int tid = threadIdx.x + threadIdx.y * blockDim.x;
    const int bid = blockIdx.x + blockIdx.y * gridDim.x;
    const int numThread = blockDim.x * blockDim.y; // Fix: Include both x and y dimensions for numThread
    const int resultID = bid * numThread + tid;
    int pos = startPos + resultID;

    if (pos < rLen) {
        int2 value = d_Rin[pos];
        int offset = value.x;
        int size = value.y;
        int startWritePos = d_sum[pos];
        char *source = (char *)d_source + offset; // Optimize: Offset source pointer directly
        char *dest = (char *)d_dest + startWritePos; // Optimize: Offset dest pointer directly

        #pragma unroll 4 // Optimize: Unroll loop for potential performance enhancement
        for (int i = 0; i < size; i++) {
            dest[i] = source[i];
        }

        value.x = startWritePos;
        d_Rin[pos] = value;
    }
}