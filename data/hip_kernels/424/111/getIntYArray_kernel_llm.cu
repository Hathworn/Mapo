#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getIntYArray_kernel(int2* d_input, int startPos, int rLen, int* d_output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    int pos = startPos + idx;
    if (pos < rLen)
    {
        d_output[pos] = d_input[pos].y;  // Directly access y-component
    }
}