#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate thread index and total number of threads
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    // Loop over target elements using strided access pattern
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Load from source and store to target with global memory access
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}