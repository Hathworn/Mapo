#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0;
    
    // Improved memory access and loop unrolling
    while (eidx < numElements) {
        sum += a[eidx] * b[eidx];
        eidx += gridDim.x * DP_BLOCKSIZE;
    }

    // Storing partial sum to shared memory
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Reduce using loop to improve warp efficiency
    for (uint stride = DP_BLOCKSIZE / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}