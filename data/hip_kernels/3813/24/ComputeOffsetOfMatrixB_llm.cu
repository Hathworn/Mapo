#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOffsetOfMatrixB(const int32_t* __restrict__ row_sum, int32_t* __restrict__ output, int32_t N) {
    // Use shared memory for storing row_sum to reduce global memory access.
    __shared__ int32_t shared_row_sum;
    if (threadIdx.x == 0) {
        shared_row_sum = -row_sum[blockIdx.x];
    }
    __syncthreads();

    // Optimize loop stride access and avoid multiple calculations.
    for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
        output[blockIdx.x * N + i] = shared_row_sum;
    }
}