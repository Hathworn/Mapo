#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global index
    if (idx < N) {  // Check if index is within bounds
        output[blockIdx.x * N + idx] = -col_sum[idx];  // Direct indexing for optimization
    }
}