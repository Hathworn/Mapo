#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use a unique thread index for seed initialization
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x; // Calculate thread index globally
    hiprand_init(seed, idx, 0, &state[idx]); // Initialize state with unique index
}