#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void reduceKernel(float *d_Result, float *d_Input, int N) {
    extern __shared__ float sdata[]; // Use shared memory for reduction
    const int tid = threadIdx.x;
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;

    for (int i = index; i < N; i += blockDim.x * gridDim.x)
        sum += d_Input[i];

    sdata[tid] = sum;
    __syncthreads();

    // Perform in-shared memory reduction
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        d_Result[blockIdx.x] = sdata[0];
    }
}