#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(const float *xs, const float *ys, float *out, const unsigned int N)
{
    // Calculate global thread index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Using stride-based loop to optimize global memory access
    for (unsigned int i = idx; i < N; i += blockDim.x * gridDim.x)
    {
        out[i] = xs[i] + ys[i];
    }
}