#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_write(char* _ptr, char* end_ptr) 
{
    // Calculate the base pointer for this block
    unsigned int* orig_ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* ptr = orig_ptr;

    // Return early if block is out of range
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Calculate the end of this block
    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);

    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Set the initial pattern at the starting pointer
    *ptr = pattern;

    // Efficiently traverse and write pattern in memory blocks
    while (true) {
        // Calculate the new pointer position using bitwise OR operation
        ptr = (unsigned int*) (((unsigned long)orig_ptr) | mask);

        // If the computed pointer goes back to the original, shift mask and continue
        if (ptr == orig_ptr) {
            mask = mask << 1;
            continue;
        }

        // Stop if the pointer has crossed the end of block
        if (ptr >= block_end) {
            break;
        }

        // Write the current pattern to the calculated position
        *ptr = pattern;

        // Update pattern and mask for the next iteration
        pattern = pattern << 1;
        mask = mask << 1;
    }
}