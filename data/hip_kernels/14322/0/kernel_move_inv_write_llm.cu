#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Exit if start of block is already out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Loop unrolling for reduced loop overhead
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = pattern;
    }
}