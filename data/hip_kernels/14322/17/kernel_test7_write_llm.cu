#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    // Compute the starting point for this block and thread
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* start_ptr = (unsigned int*)_start_ptr;

    // Terminate if pointer is out of bounds
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Unroll loop for performance
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = start_ptr[i];
    }

    return;
}