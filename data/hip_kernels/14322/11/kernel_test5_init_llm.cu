#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_init(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Exit if pointer exceeds end_ptr
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int p1 = 1;
    // Loop unrolling for efficiency
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i += 16) {
        unsigned int p2 = ~p1;

        #pragma unroll
        for (unsigned int j = 0; j < 16; j++) {
            ptr[i + j] = (j % 4 < 2) ? p1 : p2;
        }

        p1 = p1 << 1;
        // Reset p1 if zero
        if (p1 == 0) {
            p1 = 1;
        }
    }
}