#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check bounds of input pointer
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Merge copy operations to reduce loop overhead
    for (unsigned int i = 0; i < half_count; ++i) {
        if (i < half_count - 8) {
            ptr[i + 8] = ptr_mid[i];
        }
        if (i < 8) {
            ptr[i] = ptr_mid[half_count - 8 + i];
        }
        // Only perform this copy when index is valid
        ptr_mid[i] = ptr[i];
    }
}