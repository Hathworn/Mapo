#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    // Use __syncthreads() to ensure all threads reach this point before proceeding
    __syncthreads();

    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE + threadIdx.x * sizeof(unsigned int));

    // Check boundary condition for each thread
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Write pattern using each thread for its own memory slot
    *ptr = pattern;

    // Synchronize threads before returning
    __syncthreads();

    return;
}