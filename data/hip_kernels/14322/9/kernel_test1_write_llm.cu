#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Check boundaries to exit early
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }
    
    unsigned long* local_ptr = ptr;
    unsigned long* local_end_ptr = ptr + BLOCKSIZE / sizeof(unsigned long);
    
    // Loop unrolling for efficiency
    #pragma unroll
    for (; local_ptr < local_end_ptr; local_ptr++) {
        *local_ptr = (unsigned long) local_ptr;
    }

    return;
}
```
