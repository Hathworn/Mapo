#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    // Calculate starting pointer for each block
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check if the starting pointer of this block is beyond the end
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Calculate thread-specific offset for intra-block parallelism
    unsigned int thread_offset = threadIdx.x * (BLOCKSIZE / sizeof(unsigned int) / blockDim.x);
    unsigned int* local_ptr = ptr + thread_offset;

    // Calculate the number of elements each thread will handle
    unsigned int elements_per_thread = (BLOCKSIZE / sizeof(unsigned int)) / blockDim.x;

    unsigned int k = offset + thread_offset * elements_per_thread;
    unsigned int pat = pattern;

    // Loop through each element assigned to this thread
    for (unsigned int i = 0; i < elements_per_thread; i++) {
        local_ptr[i] = pat; // Write pattern to memory
        k++;
        if (k >= 32) { // Reset pattern every 32 iterations
            k = 0;
            pat = lb;
        } else {
            pat = pat << 1; // Left shift pattern
            pat |= sval; // OR with sval
        }
    }
}
```
