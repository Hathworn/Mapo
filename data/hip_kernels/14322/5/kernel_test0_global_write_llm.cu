#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    // Use unsigned long long for pointers to ensure 64-bit arithmetic
    unsigned long long* ptr = (unsigned long long*)_ptr;
    unsigned long long* end_ptr = (unsigned long long*)_end_ptr;
    unsigned long long orig_ptr_val = (unsigned long long)ptr;

    unsigned int pattern = 1;
    unsigned long long mask = 4;

    *ptr = pattern;

    // Minimize pointer arithmetic and condition checks
    while (true) {
        unsigned long long next_ptr_val = orig_ptr_val | mask;
        if (next_ptr_val >= (unsigned long long)end_ptr) {
            break; // Exit loop if next pointer exceeds the end pointer
        }

        ptr = (unsigned long long*)next_ptr_val;
        *ptr = pattern; // Write pattern to calculated pointer

        pattern <<= 1; // Efficient pattern update
        mask <<= 1;    // Efficient mask update
    }
    return;
}