#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Terminate early if out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int blockSteps = BLOCKSIZE / sizeof(unsigned int);

    // Loop unrolling for better performance
    for (unsigned int i = offset; i < blockSteps; i += MOD_SZ) {
        ptr[i] = p1;
    }

    // Reduced modulo operations
    for (unsigned int i = 0; i < blockSteps; i++) {
        if (i % MOD_SZ != offset) {
            ptr[i] = p2;
        }
    }
}