#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use thread index to perform operation on N elements if necessary.
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    if (threadIdx.x == 0) {
        // Only thread 0 performs the division to minimize overhead.
        *accuracy /= N;
    }
}