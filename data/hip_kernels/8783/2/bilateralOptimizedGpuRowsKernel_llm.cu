#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int BLOCKDIM = 16;

/**
* @brief      Calculates the Euclidean distance between two points (x0, y0) and
*             (x1, y1)
*
* @param[in]  x0    The x0 coordinate
* @param[in]  y0    The y0 coordinate
* @param[in]  x1    The x1 coordinate
* @param[in]  y1    The y1 coordinate
*
* @return     The distance between the two points
*/
__device__ inline float gaussian(float x, float mu, float sigma)
{
    return static_cast<float>(expf(-((x - mu) * (x - mu))/(2 * sigma * sigma)) / (2 * M_PI * sigma * sigma));
}

__device__ inline float distance(int x0, int y0, int x1, int y1)
{
    return static_cast<float>(sqrtf((x0 - x1) * (x0 - x1) + (y0 - y1) * (y0 - y1)));
}

__global__ void bilateralOptimizedGpuRowsKernel( float* __restrict__ inputImage, float* __restrict__ outputImage, int rows, int cols, uint32_t window, float sigmaD, float sigmaR)
{
    float filteredPixel = 0, wP = 0;
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col >= cols || row >= rows) return;

    // Cache currentPixel and use shared memory for neighbour pixels if possible
    float currentPixel = inputImage[col + row * cols];

    #pragma unroll
    for (int windowRow = 0; windowRow < window; windowRow++)
    {
        int neighbourRow = row - (window / 2) - windowRow;
        neighbourRow = max(neighbourRow, 0); // Ensure neighbourRow is within bounds

        float neighbourPixel = inputImage[col + neighbourRow * cols];

        float gR = gaussian(neighbourPixel - currentPixel, 0.0f, sigmaR);
        float gD = gaussian(distance(col, row, col, neighbourRow), 0.0f, sigmaD);

        filteredPixel += neighbourPixel * (gR * gD);
        wP += (gR * gD);
    }

    outputImage[col + row * cols] = filteredPixel / wP;
}