#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int BLOCKDIM = 16;

__device__ inline float gaussian(float x, float mu, float sigma)
{
    return expf(-((x - mu) * (x - mu)) / (2 * sigma * sigma)) / (2 * M_PI * sigma * sigma);
}

__device__ inline float distance(int x0, int y0, int x1, int y1)
{
    return sqrtf((x0 - x1) * (x0 - x1) + (y0 - y1) * (y0 - y1));
}

__global__ void bilateralNaiveGpuKernel(float* inputImage, float* outputImage, int rows, int cols, uint32_t window, float sigmaD, float sigmaR)
{
    float filteredPixel, neighbourPixel, currentPixel;
    float wP, gR, gD;
    int neighbourCol;
    int neighbourRow;
    
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Check bounds
    if (col >= cols || row >= rows) return;

    filteredPixel = 0;
    wP = 0;
    currentPixel = inputImage[col + row * cols]; // Precompute currentPixel
    
    for (int windowRow = -(window / 2); windowRow <= window / 2; ++windowRow)
    {
        for (int windowCol = -(window / 2); windowCol <= window / 2; ++windowCol)
        {
            neighbourCol = min(max(col + windowCol, 0), cols - 1);
            neighbourRow = min(max(row + windowRow, 0), rows - 1);

            neighbourPixel = inputImage[neighbourCol + neighbourRow * cols];

            // Intensity factor
            gR = gaussian(neighbourPixel - currentPixel, 0.0, sigmaR);
            // Distance factor
            gD = gaussian(distance(col, row, neighbourCol, neighbourRow), 0.0, sigmaD);

            float weight = gR * gD;
            filteredPixel += neighbourPixel * weight;
            wP += weight;
        }
    }
    if (wP > 0) // Prevent division by zero
        outputImage[col + row * cols] = filteredPixel / wP;
    else
        outputImage[col + row * cols] = currentPixel; // Preserve original pixel if no weights
}