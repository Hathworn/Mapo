#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    __shared__ float uni;
    if (threadIdx.x == 0) {
        // Load uniform value once per block
        uni = uniforms[blockIdx.x + blockOffset];
    }

    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x;

    __syncthreads();

    // Add uniform to both addresses handled by this thread
    if (address < n) {
        g_data[address] += uni;
    }
    if (address + blockDim.x < n) {
        g_data[address + blockDim.x] += uni;
    }
}