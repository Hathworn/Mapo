#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init_i32(int* vector, int value, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Use grid-stride loop for better optimization
    for (int i = idx; i < len; i += stride) {
        vector[i] = value;
    }
}