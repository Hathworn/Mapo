#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sigmoid_f32 (float* vector, float* output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds and compute sigmoid in an optimized way
    if (idx < len) {
        float tmp = vector[idx];
        float exp_tmp = expf(-fabsf(tmp)); // Use optimized exp and fabs functions
        float denom = 1.0f + exp_tmp;
        output[idx] = 0.5f * exp_tmp / denom;
    }
}