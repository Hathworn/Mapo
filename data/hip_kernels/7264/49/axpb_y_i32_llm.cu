#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpb_y_i32 (int a, int* x, int b, int* y, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        int tmp = y[idx]; // Load y[idx] to register
        tmp *= a * x[idx] + b; // Perform operation in register
        y[idx] = tmp; // Store result back to y[idx]
    }
}