#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_i32 (int* left_op, int* right_op, int* output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use loop to handle multiple elements per thread
    for (int i = idx; i < len; i += gridDim.x * blockDim.x) {
        output[i] = left_op[i] * right_op[i];
    }
}