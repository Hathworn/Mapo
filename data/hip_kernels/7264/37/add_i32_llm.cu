#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_i32 (int* left_op, int* right_op, int* output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use loop to handle cases where total threads are less than length
    for(int i = idx; i < len; i += blockDim.x * gridDim.x) {
        output[i] = left_op[i] + right_op[i];
    }
}