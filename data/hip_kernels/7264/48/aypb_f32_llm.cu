#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void aypb_f32 (float a, float* y, float b, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll loop to improve computational efficiency
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < len; i += stride) {
        y[i] = a * y[i] + b;
    }
}