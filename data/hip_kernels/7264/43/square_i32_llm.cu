#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_i32(int* __restrict__ vector, int* __restrict__ output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        // Use register memory for computation, restrict pointers for optimization
        int val = vector[idx];
        output[idx] = val * val;
    }
}