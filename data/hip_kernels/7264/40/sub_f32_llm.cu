#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sub_f32(float* left_op, float* right_op, float* output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Efficiently handle boundary condition
    if (idx < len) {
        output[idx] = left_op[idx] - right_op[idx];
    }
}