#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_f32(float* left_op, float* right_op, float* output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds before accessing arrays
    if (idx >= len) return;

    // Compute product directly
    output[idx] = left_op[idx] * right_op[idx];
}