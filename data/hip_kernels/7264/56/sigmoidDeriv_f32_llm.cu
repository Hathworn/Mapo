#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sigmoidDeriv_f32 (float* vector, float* output, int len) {
    // Use a grid-wide stride loop for improved memory access pattern
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < len; idx += gridDim.x * blockDim.x) {
        float val = vector[idx];
        float tmp = 1.0f + fabsf(val);  // Utilize the device function for absolute value
        output[idx] = -0.5f / (tmp * tmp); // Simplified computation
    }
}