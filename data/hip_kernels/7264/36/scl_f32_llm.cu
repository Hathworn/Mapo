#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scl_f32 (float* __restrict__ vector, float value, float* __restrict__ output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        // Use of __restrict__ to optimize memory access
        output[idx] = __fmaf_rn(vector[idx], value, 0.0f); // Use fused multiply-add for potential performance gain
    }
}