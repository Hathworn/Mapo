#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void xvpy_f32(float* __restrict__ x, float* __restrict__ v, float* __restrict__ y, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        // Optimize memory access by using registers
        float x_val = x[idx];
        float v_val = v[idx];
        y[idx] += x_val * v_val;
    }
}