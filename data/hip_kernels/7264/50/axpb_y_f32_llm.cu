#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpb_y_f32 (float a, float* x, float b, float* y, int len) {
    // Calculate a unique index for each thread in the grid
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the index is within the boundaries of the array
    if (idx < len) {
        float x_val = x[idx]; // Cache x[idx] to improve memory access time
        y[idx] = y[idx] * a * x_val + b; // Simplified arithmetic operation
    }
}