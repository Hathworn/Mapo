#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addValue_f32 (float* __restrict__ vector, float value, float* __restrict__ output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        // Perform addition and assign the result
        output[idx] = vector[idx] + value;
    }
}