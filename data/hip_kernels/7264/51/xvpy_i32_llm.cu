#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xvpy_i32 (int* x, int* v, int* y, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use loop unrolling to optimize for memory access
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < len; i += stride) {
        y[i] += x[i] * v[i];
    }
}