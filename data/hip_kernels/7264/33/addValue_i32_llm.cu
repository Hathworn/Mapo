#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addValue_i32 (int* vector, int value, int* output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for loop unrolling
    for (int i = idx; i < len; i += stride) { // Loop unrolling for better load balancing
        output[i] = vector[i] + value;
    }
}