#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory for efficient access
__global__ void binarize_i32 (int* vector, int threshold, int* output, int len) {
    __shared__ int sharedVector[256]; // Allocate shared memory space
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Load data into shared memory
    if (idx < len) {
        sharedVector[threadIdx.x] = vector[idx];
        __syncthreads(); // Ensure all threads have loaded their data
        // Perform computation using shared memory
        output[idx] = sharedVector[threadIdx.x] > threshold ? 1 : 0;
    }
}
```
