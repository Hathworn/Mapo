#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_f32(float* vector, float threshold, float* output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use conditional move to avoid control divergence
    float value = vector[idx];
    output[idx] = (idx < len) && (value > threshold);
}