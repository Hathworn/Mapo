#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scl_i32 (int* __restrict__ vector, int value, int* __restrict__ output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        // Use registers for intermediate calculations
        int temp = vector[idx];
        output[idx] = temp * value;
    }
}