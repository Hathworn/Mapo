#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution(float* input, int inputRows, int inputCols, int inputLd, float* kernel, int kernelRows, int kernelCols, int kernelLd, int rowStep, int colStep, float* output, int outputLd) {

    int row = (blockIdx.y * blockDim.y + threadIdx.y) * rowStep;
    int col = (blockIdx.x * blockDim.x + threadIdx.x) * colStep;

    // Ensuring threads only compute valid output elements
    if (row <= inputRows - kernelRows && col <= inputCols - kernelCols) {
        float sum = 0.0f; // Initialize sum for this output element
        for (int i = 0; i < kernelRows; i++) {
            for (int j = 0; j < kernelCols; j++) {
                // Accumulate product of kernel and corresponding input
                sum += kernel[i + j * kernelLd] * input[(row + i) + (col + j) * inputLd];
            }
        }
        output[row + col * outputLd] = sum; // Store accumulated sum in output
    }
}