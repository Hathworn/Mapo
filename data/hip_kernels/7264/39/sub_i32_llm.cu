#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sub_i32 (int* __restrict__ left_op, int* __restrict__ right_op, int* __restrict__ output, int len) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Process only if within bounds
    if (idx < len) {
        // Use registers for local variable storage
        int left_val = left_op[idx];
        int right_val = right_op[idx];
        
        // Perform subtraction
        output[idx] = left_val - right_val;
    }
}