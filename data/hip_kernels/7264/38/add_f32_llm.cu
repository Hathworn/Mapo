#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_f32(float* left_op, float* right_op, float* output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Calculate total number of threads in grid
    for (int i = idx; i < len; i += stride) { // Use loop to continue processing in strides
        output[i] = left_op[i] + right_op[i];
    }
}