#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tanh_f32(float* vector, float* output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        float tmp = vector[idx];
        // Use intrinsic function for faster computation
        output[idx] = __fdividef(tmp, 1.0f + fabsf(tmp));
    }
}