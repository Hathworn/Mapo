#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void x_avpb_py_f32(float* x, float a, float* v, float b, float* y, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use loop unrolling for potential performance gain
    if (idx < len) {
        y[idx] += x[idx] * (a * v[idx] + b);
        if (idx + blockDim.x < len) {
            y[idx + blockDim.x] += x[idx + blockDim.x] * (a * v[idx + blockDim.x] + b);
        }
    }
}