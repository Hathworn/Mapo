#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void aypb_i32 (int a, int* y, int b, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global thread index
    if (idx >= len) return; // Early exit if index is out of bounds

    // Perform the operation only for valid indices
    y[idx] = a * y[idx] + b; 
}