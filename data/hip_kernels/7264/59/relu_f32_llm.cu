#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_f32 (float* vector, float* output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll the loop for better performance
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < len; i += stride) {
        output[i] = vector[i] > 0.0f ? vector[i] : 0.0f;
    }
}