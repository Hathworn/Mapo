#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_f32(float* vector, float* output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop for better performance
    if (idx < len) {
        output[idx] = vector[idx] * vector[idx];
    }
    if (idx + blockDim.x < len) {
        output[idx + blockDim.x] = vector[idx + blockDim.x] * vector[idx + blockDim.x];
    }
    if (idx + 2 * blockDim.x < len) {
        output[idx + 2 * blockDim.x] = vector[idx + 2 * blockDim.x] * vector[idx + 2 * blockDim.x];
    }
    if (idx + 3 * blockDim.x < len) {
        output[idx + 3 * blockDim.x] = vector[idx + 3 * blockDim.x] * vector[idx + 3 * blockDim.x];
    }
}