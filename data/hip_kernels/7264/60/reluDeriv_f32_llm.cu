#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reluDeriv_f32(float* vector, float* output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use warp-level intrinsic to avoid branch divergence
    if (idx < len) {
        float val = vector[idx];
        output[idx] = __frcp_rn(__fadd_rn(1.0f, __frcp_rn(__fmul_rn(val, __fadd_rn(val, -1.0f)))));
    }
}