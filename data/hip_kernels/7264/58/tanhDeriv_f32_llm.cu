#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tanhDeriv_f32(float* vector, float* output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        // Optimize redundant operations and improve readability
        float tmp = fabsf(vector[idx]);
        float denom = 1.0f + tmp;
        output[idx] = 1.0f / (denom * denom);
    }
}