#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void customErrorCalc_f32(float* vector, float* ideal_vector, float threshold, float scaleFoff, float scaleFon, float* output, int len) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce global memory access
    __shared__ float shared_vector[256];
    
    if (idx < len) {
        shared_vector[threadIdx.x] = vector[idx];
        __syncthreads();

        float vectorValue = shared_vector[threadIdx.x];
        if (ideal_vector[idx] > threshold) {
            output[idx] = 1.0f - vectorValue;
            if (vectorValue < threshold) {
                output[idx] *= scaleFoff;
            }
        } else {
            output[idx] = vectorValue * vectorValue;
            if (vectorValue > threshold) {
                output[idx] *= scaleFon;
            }
        }
    }
}