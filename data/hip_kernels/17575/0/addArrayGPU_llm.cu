#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addArrayGPU(int* a, int* b, int* c, int n) {
    // Use blockIdx and blockDim to handle larger arrays
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}