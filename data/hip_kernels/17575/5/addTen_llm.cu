#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addTen(float* d, int count) {
    // Calculate the global thread index
    int tid = blockIdx.x * blockDim.x * blockDim.y * blockDim.z 
            + blockIdx.y * blockDim.x * blockDim.y * gridDim.x 
            + blockIdx.z * blockDim.x * blockDim.y * gridDim.x * gridDim.y 
            + threadIdx.z * blockDim.x * blockDim.y 
            + threadIdx.y * blockDim.x 
            + threadIdx.x;

    // Ensure only valid threads perform computation
    if(tid < count) {
        d[tid] += 10.0f;  // Add ten to the value at the current index
    }
}