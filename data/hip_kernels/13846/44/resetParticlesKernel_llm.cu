#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void resetParticlesKernel(float3* pos, float3* vel, float* age, float* life, int nParts)
{
    // Calculate global thread index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds and update particle properties
    if (idx < nParts) {
        pos[idx] = {0.0f, 0.0f, 0.0f};  // Use direct float assignment
        vel[idx] = {0.0f, 0.0f, 0.0f};  // Use direct float assignment
        age[idx] = 1.0f;               // Explicit float literal
        life[idx] = 1.0f;              // Explicit float literal
    }
}