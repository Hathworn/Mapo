#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogisticGrad(float* mat, float* targets, float* out_grad, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < numEls) {  // Ensure thread processes valid index
        out_grad[idx] = (targets[idx] < 0) ? 0 : (mat[idx] - targets[idx]);
    }
}