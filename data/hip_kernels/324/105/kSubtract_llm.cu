#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSubtract(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate the unique index for the thread
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop to allow multiple passes of all elements if needed
    for (unsigned int i = idx; i < numEls; i += blockDim.x * gridDim.x) {
        // Perform subtraction
        dest[i] = a[i] - b[i];
    }
}