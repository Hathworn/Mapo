#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kUpperBound(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll loop to improve memory_coalescing and performance
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        float val1 = mat1[i];
        float val2 = mat2[i];
        target[i] = val1 > val2 ? val2 : val1;
    }
}