#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kRectifyBoundingBox( float* boxes, float* width_offset, float* height_offset, float* flip, int num_images, int patch_width, int patch_height, int num_locs) {
    int loc_id = blockIdx.x * blockDim.x + threadIdx.x; // Optimize: calculate global index for location
    if (loc_id < num_locs) { // Optimize: boundary check for location index
        float *xmin_block = boxes + num_images * loc_id,
              *ymin_block = boxes + num_images * (loc_id + num_locs),
              *xmax_block = boxes + num_images * (loc_id + num_locs * 2),
              *ymax_block = boxes + num_images * (loc_id + num_locs * 3);

        int image_id = threadIdx.x; // Optimize: use threadIdx for image offset
        while (image_id < num_images) { // Optimize: loop optimization by while condition
            float xmin = (flip[image_id] > 0.5f) ? (256.0f / patch_width - xmax_block[image_id]) : xmin_block[image_id],
                  xmax = (flip[image_id] > 0.5f) ? (256.0f / patch_width - xmin_block[image_id]) : xmax_block[image_id];

            float wo = width_offset[image_id], ho = height_offset[image_id];
            xmin_block[image_id] = xmin - wo / patch_width;
            xmax_block[image_id] = xmax - wo / patch_width;

            ymin_block[image_id] = ymin_block[image_id] - ho / patch_height;
            ymax_block[image_id] = ymax_block[image_id] - ho / patch_height;

            image_id += blockDim.x; // Move to next image
        }
    }
}