#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate row and column index for the current element
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col < width) { // Only execute if within matrix width
        for (unsigned int row = 0; row < height; ++row) {
            unsigned int idx = row * width + col;
            tgtMat[idx] = mat[idx] + vec[row]; // Direct row access
        }
    }
}