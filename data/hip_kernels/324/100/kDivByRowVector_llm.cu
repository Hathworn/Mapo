#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDivByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use stride loop to improve memory access patterns
    for (unsigned int i = idx; i < width * height; i += stride) {
        unsigned int row = i / width;
        target[i] = mat[i] / vec[row];
    }
}