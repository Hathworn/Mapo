#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kExpandAndAdd(float* source, float* mat, float* indices, float* target, int width, int height, float mult, int width2) {
    // Calculate thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use shared memory to minimize global memory access
    __shared__ float sharedIndices[256]; // Assuming max blockDim.x is 256
    if (threadIdx.x < height)
        sharedIndices[threadIdx.x] = indices[threadIdx.x];

    __syncthreads();

    // Loop with stride to handle larger data
    for (unsigned int i = idx; i < width * height; i += stride) {
        // Pre-calculate position in mat for efficiency
        const int currentIdx = i / height;
        const int posIndex = sharedIndices[currentIdx];
        const int pos = height * posIndex + i % height;
        
        // Conditional check for valid position
        if (pos < height * width2) {
            target[i] = source[i] + mult * mat[pos];
        } else {
            target[i] = NAN; // Use NAN for invalid operations
        }
    }
}