#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kPow(float* mat, float pow, float* target, unsigned int len) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll the loop to reduce loop overhead
    for (; idx + 3 < len; idx += blockDim.x * gridDim.x * 4) {
        target[idx] = powf(mat[idx], pow);
        target[idx + 1] = powf(mat[idx + 1], pow);
        target[idx + 2] = powf(mat[idx + 2], pow);
        target[idx + 3] = powf(mat[idx + 3], pow);
    }

    // Handle remaining elements
    for (; idx < len; idx += blockDim.x * gridDim.x) {
        target[idx] = powf(mat[idx], pow);
    }
}