#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddScalar(float* a, float alpha, float* dest, unsigned int numEls) {
    // Calculate the global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by using a conditional statement to reduce unnecessary iterations
    if (idx < numEls) {
        dest[idx] = a[idx] + alpha;
    }
}