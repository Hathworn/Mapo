#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kFloor(float* mat, float* target, unsigned int len) {
    // Calculate flattened thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process elements that this thread is responsible for
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = floorf(mat[i]); // Use floorf for single precision float
    }
}