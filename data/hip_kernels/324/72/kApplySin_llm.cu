#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplySin(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use coalesced memory access by ensuring threads access consecutive memory addresses
    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = __sinf(mat[i]);
    }
}