#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate unique thread index
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure within bounds of the matrix
    if (x < width && y < height) {
        unsigned int index = y * width + x;
        tgtMat[index] = mat[index] + vec[y];  // Efficient vector addition using 2D indices
    }
}