#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kReadRows(float* data, float* target, int num_images, int num_modules, int num_modules_batch, int module_id_offset) {
    // Perform calculations to determine offsets outside of the loop
    int c = blockIdx.y;
    int src_module_id = module_id_offset + blockIdx.x;
    int dst_module_id = blockIdx.x;

    int src_offset = num_images * (src_module_id + c * num_modules);
    int dst_offset = num_images * (dst_module_id + c * num_modules_batch);

    // Use shared memory to reduce global memory access latency
    __shared__ float cachedData[1024];  // Adjust the size according to blockDim.x
    
    data += src_offset;
    target += dst_offset;

    for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
        // Load data into shared memory
        cachedData[threadIdx.x] = data[im];
        __syncthreads();  // Synchronize threads to ensure data integrity

        // Read from shared memory instead of global memory
        target[im] = cachedData[threadIdx.x];
        __syncthreads();  // Synchronize threads to ensure next loop iteration is consistent
    }
}