#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kContract(float *expanded_data, float* targets, int num_images, int num_input_channels, int image_size_y, int image_size_x, int num_modules_y, int num_modules_x, int kernel_size_y, int kernel_size_x, int padding_y, int padding_x, int stride_y, int stride_x, int num_modules_batch, int module_id_offset) {
    int color = blockIdx.y;
    int dst_module_id = module_id_offset + blockIdx.x;
    int module_id_x = dst_module_id % num_modules_x;
    int module_id_y = dst_module_id / num_modules_x;
    int startX = module_id_x * stride_x + padding_x;
    int startY = module_id_y * stride_y + padding_y;
    targets += num_images * image_size_x * image_size_y * color;
    expanded_data += num_images * (blockIdx.x + num_modules_batch * (kernel_size_y * kernel_size_x * color));

    for (int y = 0; y < kernel_size_y; y++) {
        int Y = startY + y;
        if (Y < 0 || Y >= image_size_y) continue;  // Early exit if Y is out of bounds
        for (int x = 0; x < kernel_size_x; x++) {
            int X = startX + x;
            if (X < 0 || X >= image_size_x) continue;  // Early exit if X is out of bounds
            long source_id = num_images * num_modules_batch * (x + kernel_size_x * y);
            long target_id = num_images * (X + image_size_x * Y);
            for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
                atomicAdd(&targets[target_id + im], expanded_data[source_id + im]);
            }
        }
    }
}