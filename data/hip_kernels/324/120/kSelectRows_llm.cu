#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSelectRows(float* source, float* target, float* indices, int nRowIs, int nCols, int nSourceRows) {
    __shared__ int sourceRowIndices[32];
    const int startTargetRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs - startTargetRowI);

    // Load row indices cooperatively and perform conditional checks
    if (tid < localNRowIs) {
        int srcIdx = int(indices[startTargetRowI + tid]);
        // Ensuring the index is within bounds
        srcIdx = (srcIdx < 0) ? (srcIdx + nSourceRows) : srcIdx;
        sourceRowIndices[tid] = (srcIdx < 0 || srcIdx >= nSourceRows) ? -1 : srcIdx;
    }
    __syncthreads();

    // Efficiently copy rows with multiple threads per row
    for (int i = 0; i < localNRowIs; i++) {
        const int targetRowI = startTargetRowI + i;
        const int sourceRowI = sourceRowIndices[i];
        if (sourceRowI != -1) {  // Skip if invalid index
            for (int colI = tid; colI < nCols; colI += 32) {
                target[targetRowI * nCols + colI] = source[sourceRowI * nCols + colI];
            }
        }
    }
}