#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddMultSign(float* a, float* b, unsigned int numEls, float mult) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x; // Calculate stride once

    for (unsigned int i = idx; i < numEls; i += stride) {
        float b_value = b[i]; // Cache b[i] value
        float addition = (b_value > 0) ? mult : ((b_value < 0) ? -mult : 0); // Determine addition
        a[i] += addition; // Directly update a[i]
    }
}