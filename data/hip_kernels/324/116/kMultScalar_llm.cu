#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultScalar(float* mat, float alpha, float* dest, unsigned int len, float scale_targets) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Unifying loops for better readability and potential performance improvements
    for (unsigned int i = idx; i < len; i += stride) {
        dest[i] = (scale_targets == 0 ? 0 : scale_targets * dest[i]) + alpha * mat[i];
    }
}