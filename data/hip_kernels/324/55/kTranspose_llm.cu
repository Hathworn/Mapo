#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTranspose(float *odata, float *idata, int width, int height) {
    __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE+1];

    // calculate indexes for reading the matrix tile
    unsigned int xIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;

    // load matrix tile into shared memory
    if((xIndex < width) && (yIndex < height)) {
        unsigned int index_in = yIndex * width + xIndex;
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    __syncthreads();

    // calculate indexes for writing the transposed matrix tile
    xIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
    yIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

    // store transposed tile into output global memory
    if((xIndex < height) && (yIndex < width)) {
        unsigned int index_out = yIndex * height + xIndex;
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}