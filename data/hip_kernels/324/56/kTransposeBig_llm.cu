#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTransposeBig(float *odata, float *idata, int height, int width) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Calculate transposed index more efficiently
    for (unsigned int i = idx; i < width * height; i += numThreads) {
        unsigned int r = i % width;
        unsigned int c = i / width;
        odata[c * width + r] = idata[i];
    }
}