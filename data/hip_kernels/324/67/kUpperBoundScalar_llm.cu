#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kUpperBoundScalar(float* mat, float val, float* target, unsigned int len) {
    // Calculate global thread index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop through data using stride to ensure all elements are processed
    while (idx < len) {
        target[idx] = mat[idx] > val ? val : mat[idx];
        idx += blockDim.x * gridDim.x;  // Move to the next data chunk
    }
}