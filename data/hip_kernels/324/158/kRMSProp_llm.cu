#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kRMSProp(float *history, float *grad, float factor, int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float shared_history[256];  // Use shared memory for better performance
    __shared__ float shared_grad[256];

    const unsigned int local_idx = threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    
    for (unsigned int i = idx; i < len; i += numThreads) {
        if (i < len) {
            shared_history[local_idx] = history[i];  // Load data into shared memory
            shared_grad[local_idx] = grad[i];
            __syncthreads();  // Ensure all threads have loaded their data

            // Perform computation with shared memory
            shared_history[local_idx] = sqrt(factor * shared_history[local_idx] * shared_history[local_idx] + 
                                             (1-factor) * shared_grad[local_idx] * shared_grad[local_idx]);
            __syncthreads();  // Ensure computation is complete before writing back

            history[i] = shared_history[local_idx];  // Write back results to global memory
        }
    }
}
```
