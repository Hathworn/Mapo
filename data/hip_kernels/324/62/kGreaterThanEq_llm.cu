#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kGreaterThanEq(float* mat1, float* mat2, float* target, unsigned int len) {
    // Calculate global index for the current thread
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride to ensure all elements are covered by the threads
    for (unsigned int i = idx; i < len; i += gridDim.x * blockDim.x) {
        // Perform the comparison and store it directly
        target[i] = mat1[i] >= mat2[i];
    }
}