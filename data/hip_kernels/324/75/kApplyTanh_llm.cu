#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplyTanh(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Utilize more efficient memory access pattern
    for (unsigned int i = idx; i < len; i += stride) {
        float mat_i = mat[i];
        // Use fast mathematic intrinsic for exp
        float exp2x = __expf(2.0f * mat_i);
        target[i] = 1.0f - 2.0f / (exp2x + 1.0f);
    }
}