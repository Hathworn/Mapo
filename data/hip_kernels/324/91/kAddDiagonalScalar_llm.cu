#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddDiagonalScalar(float* mat, float val, float* tgtMat, unsigned int width) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int step = blockDim.x * gridDim.x;
    if (idx < width) { // Check if idx is within bounds
        for (unsigned int i = idx; i < width; i += step) {
            tgtMat[width * i + i] = mat[width * i + i] + val;
        }
    }
}