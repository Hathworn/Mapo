#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivideScalar(float* __restrict__ mat, float alpha, float* __restrict__ dest, unsigned int len) {
    // Calculate global thread index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unrolling and using fewer threads for data access optimization
    for (; idx < len; idx += blockDim.x * gridDim.x) {
        dest[idx] = mat[idx] / alpha;
    }
}