#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplyAbs(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single loop iteration to optimize performance
    if (idx < len) {
        target[idx] = fabsf(mat[idx]); // Utilize fast math function for absolute value
    }
}