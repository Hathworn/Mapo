#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kHingeQuadraticRowMajor(float* __restrict__ mat, float* __restrict__ labels, float* __restrict__ target, unsigned int width, unsigned int height, float margin) {
    int image_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (image_id < height) {
        mat += image_id;
        target += image_id;
        const int correct_label = static_cast<int>(labels[image_id]);
        const float correct_label_score = mat[correct_label * height];
        float sum = 0;

        // Unroll the loop using manual loop unrolling for performance gain.
        unsigned int i;
        for (i = 0; i + 4 <= width; i += 4) {
            #pragma unroll
            for (int j = 0; j < 4; ++j) { // Inner loop for unrolling step
                float diff = margin + mat[(i + j) * height] - correct_label_score;
                float grad = (diff > 0) ? diff : 0;
                target[(i + j) * height] = ((i + j) == correct_label) ? 0 : grad;
                sum += ((i + j) == correct_label) ? 0 : grad;
            }
        }

        // Handle remaining elements that were not part of the loop unrolling
        for (; i < width; i++) {
            float diff = margin + mat[i * height] - correct_label_score;
            float grad = (diff > 0) ? diff : 0;
            target[i * height] = (i == correct_label) ? 0 : grad;
            sum += (i == correct_label) ? 0 : grad;
        }

        target[correct_label * height] = -sum;
    }
}