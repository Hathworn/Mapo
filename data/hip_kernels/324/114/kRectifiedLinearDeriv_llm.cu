#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kRectifiedLinearDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll the loop to increase instruction-level parallelism
    unsigned int stride = blockDim.x * gridDim.x;
    for (unsigned int i = idx; i < numEls; i += stride) {
        float valB = b[i];  // Cache b[i] in register to reduce memory access
        dest[i] = a[i] * (valB > 0 ? 1 : 0);
    }
}