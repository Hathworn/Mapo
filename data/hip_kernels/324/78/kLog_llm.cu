#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLog(float* mat, float* target, unsigned int len, float tiny) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {  // Ensure threads process only valid indices
        target[idx] = __logf(mat[idx] + tiny);
    }
}