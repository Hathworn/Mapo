#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSwapColumns(float* source, float* target, float* indices1, float* indices2, int cols, int width, int height) {
    // Calculate flattened thread index for global memory access
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Pre-calculate variables outside of the loop for reusability
    float temp;
    unsigned int column, row, source_pos, target_pos;

    // Loop over matrix elements within the column
    for (unsigned int i = idx; i < height * cols; i += numThreads) {
        column = i / height;
        row = i % height;
        
        // Calculate source and target positions based on column indices
        source_pos = height * static_cast<int>(indices1[column]) + row;
        target_pos = height * static_cast<int>(indices2[column]) + row;

        // Perform the swap operation
        temp = source[source_pos];
        source[source_pos] = target[target_pos];
        target[target_pos] = temp;
    }
}