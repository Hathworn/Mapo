#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSign(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Loop over the data with stride of number of threads
    for (unsigned int i = idx; i < len; i += numThreads) {
        // Use ternary operator to directly assign the sign
        target[i] = (mat[i] > 0) - (mat[i] < 0);
    }
}