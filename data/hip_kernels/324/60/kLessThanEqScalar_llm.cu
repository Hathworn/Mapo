#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLessThanEqScalar(float* mat, float val, float* target, unsigned int len) {
// Calculate the global index
const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
if (idx < len) { // Ensure the thread processes only if inside the bounds
    target[idx] = mat[idx] <= val;
}
}