#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTanhDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index is within bounds
    if (idx < numEls) {
        dest[idx] = a[idx] * (1.0f + b[idx]) * (1.0f - b[idx]);
    }
}