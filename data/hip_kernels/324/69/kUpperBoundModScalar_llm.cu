#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kUpperBoundModScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll the loop for better performance
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += stride) {
        float element = mat[i];
        // Use ternary conditions to directly assign the value to target
        target[i] = element > val ? val : (element < -val ? -val : element);
    }
}