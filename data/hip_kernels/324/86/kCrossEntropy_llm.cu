#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCrossEntropy(float* mat, float* p, float* target, unsigned int len, float tiny) {
    // Calculate the global index once instead of recalculating in each iteration
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll the loop for improved efficiency
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = -mat[i] * __logf(p[i] + tiny); // Use registers for improved performance
    }
}