#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y; // Optimize for 2D grid/block
    const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < height && col < width) { // Use boundary checks
        const unsigned int idx = row * width + col;
        tgtMat[idx] = mat[idx] / vec[row]; // Use row directly for vec index
    }
}
```
