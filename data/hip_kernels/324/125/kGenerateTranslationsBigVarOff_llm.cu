#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kGenerateTranslationsBigVarOff(float* source, float* target, float* off_x_arr, float* off_y_arr, int source_w, int target_w, int num_channels) {
    const unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x; // Optimize index calculation for block
    const unsigned int stride = blockDim.x * gridDim.x; // Calculate stride for inter-block jump

    int target_x, target_y;
    int pad = (source_w - target_w) / 2;
    int target_tile_size = target_w * target_w;
    int source_tile_size = source_w * source_w;

    int off_x = off_x_arr[blockIdx.x];
    int off_y = off_y_arr[blockIdx.x];
    int target_off = blockIdx.x * target_tile_size;
    int source_off = blockIdx.x * source_tile_size + (pad + off_x) * source_w + (pad + off_y);

    for (unsigned int target_ind = idx; target_ind < target_tile_size; target_ind += stride) { // Use stride for loop
        target_x = target_ind / target_w;
        target_y = target_ind % target_w; // Optimize modulus operation

        int target_base_idx = num_channels * (target_off + target_x * target_w + target_y); // Precompute base target index
        int source_base_idx = num_channels * (source_off + target_x * source_w + target_y); // Precompute base source index

        for (unsigned int ch = 0; ch < num_channels; ch++) {
            target[target_base_idx + ch] = source[source_base_idx + ch]; // Use precomputed indices
        }
    }
}