#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAssignScalar(float* dest, float alpha, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {  // Ensure index is within bounds
        dest[idx] = alpha;  // Assign value
    }
}
```
