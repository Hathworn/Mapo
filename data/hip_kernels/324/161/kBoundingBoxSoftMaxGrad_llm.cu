#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBoundingBoxSoftMaxGrad( 
    float* __restrict__ mat, 
    int* __restrict__ bbox, 
    int* __restrict__ label, 
    int* __restrict__ seg, 
    float* __restrict__ indices, 
    float* __restrict__ width_offset, 
    float* __restrict__ height_offset, 
    int size, 
    int width, 
    int height, 
    int depth, 
    float scale_width, 
    float scale_height, 
    float* __restrict__ grad) 
{
    const unsigned int len = width * height * depth * size;
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Use register to cache frequently used variables
    __shared__ float shared_bbox[1024];
    
    for (unsigned int i = idx; i < len; i += numThreads) {
        int ind = i;
        const int image_id = ind % size; ind /= size;
        const float source_x = scale_width * (ind % width); ind /= width;
        const float source_y = scale_height * (ind % height); ind /= height;
        const int source_depth = ind % depth;
        const int src_image_id = (int) indices[image_id];

        // Move unnecessary computations outside loop
        const int start = seg[src_image_id];
        const int end = seg[src_image_id + 1];
        int num_bboxes = 0;
        int num_bboxes_of_this_depth = 0;

        for (int box_id = start; box_id < end; box_id++) {
            if (threadIdx.x < 4) shared_bbox[threadIdx.x * (end-start) + box_id - start] = bbox[(box_id << 2) + threadIdx.x];
            __syncthreads(); // Ensure all threads have loaded the relevant data
            
            const int x1 = shared_bbox[(box_id - start) * 4] - width_offset[image_id];
            const int y1 = shared_bbox[(box_id - start) * 4 + 1] - height_offset[image_id];
            const int x2 = shared_bbox[(box_id - start) * 4 + 2] - width_offset[image_id];
            const int y2 = shared_bbox[(box_id - start) * 4 + 3] - height_offset[image_id];
            
            const int inside = (source_x >= x1 && source_x <= x2 && source_y >= y1 && source_y <= y2) ? 1:0;
            num_bboxes += inside;
            num_bboxes_of_this_depth += (inside == 1 && label[box_id] == source_depth) ? 1 : 0;
        }

        grad[i] = mat[i] - ((num_bboxes > 0) ? ((float)num_bboxes_of_this_depth / num_bboxes) : (source_depth == 0 ? 1 : 0));
    }
}