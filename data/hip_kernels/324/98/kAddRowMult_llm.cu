#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddRowMult(float* mat, float* vec, float* tgtMat, float mult, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int row = idx / width; // Calculate row index
    const unsigned int col = idx % width; // Calculate column index
    if (row < height) { // Ensure index is within bounds
        tgtMat[idx] = mat[idx] + mult * vec[row];
    }
}