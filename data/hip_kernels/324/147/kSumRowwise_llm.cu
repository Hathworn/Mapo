#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSumRowwise(float* mat, float* target, unsigned int width, unsigned int height, float mult, float p) {
    extern __shared__ float sum_vals[];
    const int row = blockIdx.x * blockDim.x + threadIdx.x; // Simplify row calculation

    if (row < height) {
        float sum = 0;
        for (unsigned int i = row; i < width * height; i += height) { // Improve memory access pattern
            sum += mat[i];
        }
        target[row] = p * target[row] + mult * sum; // Remove __syncthreads() which is redundant here
    }
}