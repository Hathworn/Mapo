#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCrossEntropyBernoulli(float* mat, float* p, float* target, unsigned int len, float tiny) {
    // Calculate unique global thread ID
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure each thread processes multiple elements for optimal occupancy and performance
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        float mat_val = mat[i];
        float p_val = p[i];
        target[i] = -mat_val * __logf(p_val + tiny) - (1.0f - mat_val) * __logf(1.0f - p_val + tiny);
    }
}