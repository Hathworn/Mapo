#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSoftMaxCrossEntropy(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float tiny) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x; // Calculate stride outside of loop
    for (unsigned int i = idx; i < width; i += stride) {
        int labelIndex = static_cast<int>(labels[i]); // Cache label index
        float value = mat[height * i + labelIndex]; // Cache matrix value
        target[i] = -__logf(value + tiny); // Simplify calculation
    }
}