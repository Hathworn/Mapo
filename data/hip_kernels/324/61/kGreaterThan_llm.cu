#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kGreaterThan(float* mat1, float* mat2, float* target, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize by ensuring thread index is within bounds
    if (idx < len) {
        target[idx] = mat1[idx] > mat2[idx];
    }
}