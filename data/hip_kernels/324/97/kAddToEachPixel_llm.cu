#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddToEachPixel(float* mat1, float* mat2, float* tgtMat, float mult, unsigned int width, unsigned int height, unsigned int num_pix) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    const unsigned int total_size = width * height;

    // Use a standard for loop with a stride to improve warp efficiency
    for (unsigned int i = idx; i < total_size; i += numThreads) {
        unsigned int row = i / width;
        unsigned int col = i % width;
        unsigned int offset = col + width * (row / num_pix); // Calculate exact offset
        tgtMat[i] = mat1[i] + mult * mat2[offset];
    }
}