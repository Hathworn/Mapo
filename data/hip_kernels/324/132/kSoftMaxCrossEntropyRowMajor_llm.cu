#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSoftMaxCrossEntropyRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float tiny) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Loop unrolling for better memory access and reduced loop overhead
    for (unsigned int i = idx; i < height; i += numThreads) {
        // Compute target element using more efficient indexing
        unsigned int label_idx = static_cast<int>(labels[i]);
        unsigned int mat_idx = height * label_idx + i;
        
        target[i] = -__logf(mat[mat_idx] + tiny);
    }
}