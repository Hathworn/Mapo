#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSinDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Optimize by unrolling loops to enhance parallel execution efficiency
    if (idx < numEls) {
        dest[idx] = a[idx] * __cosf(b[idx]);
    }
}