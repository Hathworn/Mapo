#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddColMult(float* mat, float* vec, float* tgtMat, float mult, unsigned int width, unsigned int height) {
    // Calculate global index
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread does not access out-of-bounds memory
    if (i < width * height) {
        // Perform multiplication and addition
        tgtMat[i] = mat[i] + mult * vec[i % height];
    }
}