#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLowerBound(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Utilize shared memory to reduce global memory accesses
    __shared__ float s_mat1[256];
    __shared__ float s_mat2[256];
    for (unsigned int i = idx; i < len; i += stride) {
        s_mat1[threadIdx.x] = mat1[i];
        s_mat2[threadIdx.x] = mat2[i];
        __syncthreads(); // Ensure all threads have loaded their elements

        // Compute target using shared memory
        target[i] = s_mat1[threadIdx.x] < s_mat2[threadIdx.x] ? s_mat2[threadIdx.x] : s_mat1[threadIdx.x];
        __syncthreads(); // Ensure computation is complete before next iteration
    }
}