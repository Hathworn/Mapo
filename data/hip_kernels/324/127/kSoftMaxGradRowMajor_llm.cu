#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSoftMaxGradRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;
    
    // Optimization: Use shared memory for frequently accessed values
    extern __shared__ float sharedLabels[];
    
    unsigned int row = idx / width;
    
    if (threadIdx.x < height) {
        sharedLabels[threadIdx.x] = labels[threadIdx.x];
    }
    __syncthreads();
   
    for (unsigned int i = idx; i < width * height; i += numThreads) {
        unsigned int rowIndex = i / width;
        unsigned int colIndex = i % width;
        
        // Optimization: access shared memory for labels
        target[i] = mat[i] - ((int)sharedLabels[rowIndex] == colIndex ? 1 : 0);
    }
}