#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kExpand(float *images, float* targets, int num_images, int num_input_channels, int image_size_y, int image_size_x, int num_modules_y, int num_modules_x, int kernel_size_y, int kernel_size_x, int padding_y, int padding_x, int stride_y, int stride_x, int num_modules_batch, int module_id_offset) {
    int color = blockIdx.y;
    int src_module_id = module_id_offset + blockIdx.x;
    int dst_module_id = blockIdx.x;

    int module_id_x = src_module_id % num_modules_x;
    int module_id_y = src_module_id / num_modules_x;
    int startX = module_id_x * stride_x + padding_x;
    int startY = module_id_y * stride_y + padding_y;

    long target_id, source_id;
    images += num_images * image_size_x * image_size_y * color;
    targets += num_images * (dst_module_id + num_modules_batch * (kernel_size_y * kernel_size_x * color));

    for (int y = threadIdx.y; y < kernel_size_y; y += blockDim.y) {
        int Y = startY + y;
        for (int x = threadIdx.x; x < kernel_size_x; x += blockDim.x) {
            int X = startX + x;
            target_id = num_images * num_modules_batch * (x + kernel_size_x * y);
            source_id = num_images * (X + image_size_x * Y);
            if (X < 0 || X >= image_size_x || Y < 0 || Y >= image_size_y) {
                for (int im = threadIdx.z; im < num_images; im += blockDim.z) {
                    targets[target_id + im] = 0; // Assign zero for out-of-bounds
                }
            } else {
                for (int im = threadIdx.z; im < num_images; im += blockDim.z) {
                    targets[target_id + im] = images[source_id + im]; // Copy image data within bounds
                }
            }
            __syncthreads(); // Ensure all threads have completed their computations before proceeding
        }
    }
}