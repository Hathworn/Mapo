#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultDiagonalScalar(float* mat, float val, float* tgtMat, unsigned int width) {
    // Calculate thread index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds and operate only on diagonal elements
    if (i < width) {
        unsigned int idx = width * i + i;
        tgtMat[idx] = mat[idx] * val;
    }
}