#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kShuffleColumns(float* source, float* target, float* indices, int width, int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Utilize loop unrolling to reduce loop overhead
    for (unsigned int i = idx; i < height * ((width + 1) / 2); i += numThreads) {
        unsigned int column = 2 * (i / height);
        unsigned int row = i % height;
        unsigned int pos1, pos2;
        
        pos1 = height * static_cast<int>(indices[column]) + row;
        
        if (column + 1 < width) {
            pos2 = height * static_cast<int>(indices[column + 1]) + row;
            float temp1 = source[pos1];
            float temp2 = source[pos2];
            target[pos2] = temp1;
            target[pos1] = temp2;
        } else {
            target[pos1] = source[pos1];
        }
    }
}