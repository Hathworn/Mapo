#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kGreaterThanScalar(float* mat, float val, float* target, unsigned int len) {
    // Calculate the global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure the thread is within array bounds
    if (idx < len) {
        target[idx] = mat[idx] > val;
    }
}