#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetSelectedRows(float* target, float* source, float* indices, int nRowIs, int nCols, int nTargetRows){
    __shared__ int targetRowIndices[32];
    const int startSourceRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs-startSourceRowI);

    // Load row indices cooperatively with boundary checks
    if (tid < localNRowIs) {
        int index = int(indices[startSourceRowI + tid]);
        if (index < 0) index += nTargetRows;
        targetRowIndices[tid] = (index < 0 || index >= nTargetRows) ? -1 : index;
    }
    __syncthreads();

    // Copy rows efficiently using shared memory indices
    for (int i = 0; i < localNRowIs; i++) {
        int targetRowI = targetRowIndices[i];
        if (targetRowI != -1) {
            for (int colI = tid; colI < nCols; colI += 32) {
                target[targetRowI * nCols + colI] = source[(startSourceRowI + i) * nCols + colI];
            }
        }
    }
}