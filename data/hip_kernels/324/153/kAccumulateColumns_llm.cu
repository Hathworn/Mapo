#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAccumulateColumns(float* mat, float* indices, float* target, int mat_width, int target_width, int height, float mult, int avg) {
    const int row = blockIdx.y * gridDim.x + blockIdx.x;
    const int column = threadIdx.x;
    
    // Check bounds outside the loop for efficiency
    if (row < height && column < target_width) {
        float cur_sum = 0.0f;
        unsigned int count = 0;

        // Use shared memory for fast access if needed
        for (unsigned int i = 0; i < mat_width; i++) {
            int idx_match = __float2int_rz(indices[i]) == column;
            count += idx_match;  // Increment count only if index matches
            cur_sum += idx_match * mat[row + i * height]; // Accumulate sum only if index matches
        }

        // Avoid division by zero by checking count > 0
        target[row + height * column] = mult * cur_sum / ((avg == 1 && count > 0) ? count : 1);
    }
}