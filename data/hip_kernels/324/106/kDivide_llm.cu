#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDivide(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unroll the loop to process multiple elements per thread, improving performance
    while (idx < numEls) {
        dest[idx] = a[idx] / b[idx];
        idx += blockDim.x * gridDim.x;
    }
}