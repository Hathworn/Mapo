#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kBoundingBoxLogisticGrad(float* mat, int* bbox, int* label, int* seg, float* indices, float* width_offset, float* height_offset, int size, int width, int height, int depth, float scale_width, float scale_height, float* grad) {
    const int image_id = threadIdx.x;
    const int col = blockIdx.x;
    const int row = blockIdx.y;
    const int color = blockIdx.z;

    if (col >= width || row >= height || image_id >= size || color >= depth) return;  // Out-of-bounds check

    int src_image_id = __ldg(&indices[image_id]);
    int src_col = __float2int_rd(scale_width * col);
    int src_row = __float2int_rd(scale_height * row);

    int start = __ldg(&seg[src_image_id]);
    int end = __ldg(&seg[src_image_id + 1]);

    int num_bboxes = 0, num_bboxes_of_this_depth = 0, num_bboxes_of_this_depth_inside = 0;

    for (int box_id = start; box_id < end; box_id++) {
        int l = __ldg(&label[box_id]);
        int x1 = __ldg(&bbox[box_id << 2]) - __ldg(&width_offset[image_id]);
        int y1 = __ldg(&bbox[(box_id << 2) + 1]) - __ldg(&height_offset[image_id]);
        int x2 = __ldg(&bbox[(box_id << 2) + 2]) - __ldg(&width_offset[image_id]);
        int y2 = __ldg(&bbox[(box_id << 2) + 3]) - __ldg(&height_offset[image_id]);

        int inside = (src_col >= x1 && src_col <= x2 && src_row >= y1 && src_row <= y2) ? 1 : 0;
        num_bboxes += inside;
        num_bboxes_of_this_depth += (l == color);
        num_bboxes_of_this_depth_inside += (inside && l == color);
    }

    unsigned long i = image_id + size * (col + width * (row + height * color));

    if (num_bboxes > 0) {
        grad[i] = (num_bboxes_of_this_depth_inside > 0) ? (mat[i] - 1) : 0;
    } else {
        grad[i] = (num_bboxes_of_this_depth > 0) ? mat[i] : 0;
    }
}