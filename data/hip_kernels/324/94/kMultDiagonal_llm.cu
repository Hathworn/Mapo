#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultDiagonal(float* mat, float* vec, float* tgtMat, unsigned int width) {
    // Calculate the global index of the thread
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within the matrix bounds
    if (idx < width) {
        tgtMat[width * idx + idx] = mat[width * idx + idx] * vec[idx];
    }
}