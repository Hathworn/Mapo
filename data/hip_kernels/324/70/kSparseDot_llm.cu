#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSparseDot(int m, int n, int k, float *data, int* indptr, int* indices, float *dense_data, float* target, float beta, float alpha) {
    const unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < m && col < n) {
        const int start = indptr[row];
        const int end = indptr[row + 1];
        float sum = 0.f;

        // Unroll inner loop for better performance
        for (int i = start; i < end; i += 4) {
            sum += data[i] * dense_data[col * k + indices[i]];
            if (i + 1 < end) sum += data[i + 1] * dense_data[col * k + indices[i + 1]];
            if (i + 2 < end) sum += data[i + 2] * dense_data[col * k + indices[i + 2]];
            if (i + 3 < end) sum += data[i + 3] * dense_data[col * k + indices[i + 3]];
        }

        const int pos = col * m + row;
        // Combined beta condition using ternary operator
        target[pos] = alpha * sum + beta * target[pos] * (beta != 0);
    }
}