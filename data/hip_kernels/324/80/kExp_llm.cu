#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kExp(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) { // Check bounds to avoid unnecessary calculations
        target[idx] = __expf(mat[idx]); // Use idx directly since it's within bounds
    }
}