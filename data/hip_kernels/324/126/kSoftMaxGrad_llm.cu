#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSoftMaxGrad(float* mat, float* labels, float* target, unsigned int width, unsigned int height) {
    // Use shared memory to reduce global memory access 
    extern __shared__ float shared_labels[];
    
    const unsigned int tid = threadIdx.x;
    const unsigned int idx = blockIdx.x * blockDim.x + tid;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    if (tid < height) {
        shared_labels[tid] = labels[tid];
    }
    __syncthreads();

    for (unsigned int i = idx; i < width * height; i += numThreads) {
        int labelIdx = i / height; // Calculate the label index
        int correspondingLabel = (int)shared_labels[labelIdx];
        target[i] = mat[i] - (correspondingLabel == i % height ? 1.0f : 0.0f);
    }
}
```
