#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLessThanEq(float* mat1, float* mat2, float* target, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (idx < len) {
        target[idx] = mat1[idx] <= mat2[idx];
    }
}