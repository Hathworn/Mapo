#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kRectifiedLinearSmoothDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop by processing multiple elements per thread to improve memory access pattern
    for (unsigned int i = idx; i < numEls; i += blockDim.x * gridDim.x) {
        float bVal = b[i];
        dest[i] = a[i] * (1 - __expf(-bVal));
    }
}