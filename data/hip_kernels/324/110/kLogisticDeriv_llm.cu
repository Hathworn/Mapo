#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogisticDeriv(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread ID
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we don't go out of bounds
    if (idx < numEls) {
        dest[idx] = a[idx] * b[idx] * (1.0 - b[idx]);
    }
}