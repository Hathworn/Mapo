#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLessThan(float* mat1, float* mat2, float* target, unsigned int len) {
    // Calculate the index based on block and thread indices
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize the loop by removing unnecessary index calculations
    if (idx < len) {
        target[idx] = mat1[idx] < mat2[idx];
    }
}