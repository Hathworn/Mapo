#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kExpand(float* source, float* indices, float* target, int height, int width, int target_width) {
    // Calculate global thread index and total number of threads
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Iterate with stride pattern to improve coalesced memory access
    for (unsigned int i = idx; i < target_width * height; i += numThreads) {
        const int index = static_cast<int>(indices[i / height]);
        const int pos = height * index + i % height;

        // Use ternary operator for improved readability
        target[i] = (pos < height * width) ? source[pos] : __int_as_float(0x7f800000);  // Use __int_as_float for infinity
    }
}