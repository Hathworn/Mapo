#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLessThanScalar(float* mat, float val, float* target, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds and operate directly on each index
    if (idx < len) {
        target[idx] = mat[idx] < val;
    }
}