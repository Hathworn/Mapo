#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAdagrad(float *history, float *grad, float delta, int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory for reduction if applicable, else direct index
    if (idx < len) {
        float curr_norm = history[idx] - delta;
        history[idx] = delta + sqrt(curr_norm * curr_norm + grad[idx] * grad[idx]);
    }
}