#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSquashRelu(float* mat, float* target, unsigned int len, float lambda) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Loop unrolling to improve warp execution efficiency
    for (unsigned int i = idx; i < len; i += numThreads * 4) {
        if (i < len) target[i] = 2 / (1 + __expf(-lambda * mat[i])) - 1;
        if (i + numThreads < len) target[i + numThreads] = 2 / (1 + __expf(-lambda * mat[i + numThreads])) - 1;
        if (i + 2 * numThreads < len) target[i + 2 * numThreads] = 2 / (1 + __expf(-lambda * mat[i + 2 * numThreads])) - 1;
        if (i + 3 * numThreads < len) target[i + 3 * numThreads] = 2 / (1 + __expf(-lambda * mat[i + 3 * numThreads])) - 1;
    }
}