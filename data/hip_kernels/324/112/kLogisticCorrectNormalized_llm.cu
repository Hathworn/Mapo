#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogisticCorrectNormalized(float* mat, float* targets, float* out, unsigned int height, unsigned int width) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < height) {
        float correct = 0;
        float total = 0;
        for (int i = idx; i < width * height; i += height) {
            // Use registers for frequently accessed values
            float p = mat[i];
            float t = targets[i];
            
            // Simplify condition checks to reduce branch divergence
            if (t >= 0) {
                correct += ((t >= 0.5) == (p >= 0.5)) ? 1 : 0;
                total += 1;
            }
        }
        
        // No need for __syncthreads() within a single thread loop
        out[idx] = (total > 0) ? (correct / total) : 0;
    }
}