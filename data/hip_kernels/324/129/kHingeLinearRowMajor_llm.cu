#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kHingeLinearRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float margin) {
    int image_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (image_id < height) {
        mat += image_id;
        target += image_id;
        const int correct_label = (int)labels[image_id];
        const float correct_label_score = mat[correct_label * height];
        float sum = 0;

        // Loop unrolling: process four elements at a time
        for (unsigned int i = 0; i < width / 4 * 4; i += 4) {
            float diff0 = margin + mat[i * height] - correct_label_score;
            float diff1 = margin + mat[(i + 1) * height] - correct_label_score;
            float diff2 = margin + mat[(i + 2) * height] - correct_label_score;
            float diff3 = margin + mat[(i + 3) * height] - correct_label_score;

            float grad0 = (diff0 > 0) ? 1 : 0;
            float grad1 = (diff1 > 0) ? 1 : 0;
            float grad2 = (diff2 > 0) ? 1 : 0;
            float grad3 = (diff3 > 0) ? 1 : 0;

            if (i != correct_label) {
                target[i * height] = grad0;
                sum += grad0;
            }
            if (i + 1 != correct_label) {
                target[(i + 1) * height] = grad1;
                sum += grad1;
            }
            if (i + 2 != correct_label) {
                target[(i + 2) * height] = grad2;
                sum += grad2;
            }
            if (i + 3 != correct_label) {
                target[(i + 3) * height] = grad3;
                sum += grad3;
            }
        }

        // Handle the remaining elements
        for (unsigned int i = width / 4 * 4; i < width; i++) {
            float diff = margin + mat[i * height] - correct_label_score;
            float grad = (diff > 0) ? 1 : 0;
            target[i * height] = (i == correct_label) ? 0 : grad;
            sum += (i == correct_label) ? 0 : grad;
        }

        target[correct_label * height] = -sum;
    }
}