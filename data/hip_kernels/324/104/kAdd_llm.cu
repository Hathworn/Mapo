#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread does not access out-of-bounds elements
    if (idx < numEls) {
        dest[idx] = a[idx] + b[idx];
    }
}