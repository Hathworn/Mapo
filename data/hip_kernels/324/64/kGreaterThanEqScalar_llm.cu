#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kGreaterThanEqScalar(float* mat, float val, float* target, unsigned int len) {
    // Calculate linear index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is within bounds
    if (idx < len) {
        target[idx] = mat[idx] >= val;
    }
}