#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate the global index
    const unsigned int globalIdx = row * width + col;
    
    // Ensure within bounds
    if (col < width && row < height) {
        tgtMat[globalIdx] = mat[globalIdx] * vec[row];
    }
}