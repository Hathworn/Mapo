#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSoftMaxGradCLS(float* mat, int* labels, float* indices, float* target, unsigned int width, unsigned int height) {
    // Calculate flattened thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Launch one thread for each element
    if (idx < width * height) {
        // Determine column and row
        unsigned int col = idx / height;
        unsigned int row = idx % height;
        // Compute target value
        target[idx] = mat[idx] - (labels[(int)indices[row]] == col ? 1.0f : 0.0f);
    }
}