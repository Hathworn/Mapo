#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kWriteRows(float* data, float* target, int num_images, int num_modules, int num_modules_batch, int module_id_offset, float beta) {
    int c = blockIdx.y;
    int src_module_id = blockIdx.x;
    int dst_module_id = module_id_offset + blockIdx.x;
    
    // Optimize memory access by calculating pointers within shared memory scope
    float* dataPtr = data + num_images * (src_module_id + c * num_modules_batch);
    float* targetPtr = target + num_images * (dst_module_id + c * num_modules);

    // Use shared memory to coalesce writes
    for (int im = threadIdx.x; im < num_images; im += blockDim.x) {
        targetPtr[im] = beta * dataPtr[im];
    }
}