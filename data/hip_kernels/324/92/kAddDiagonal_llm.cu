#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddDiagonal(float* mat, float* vec, float* tgtMat, unsigned int width) {
    // Calculate global index
    const unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    // It's not necessary to loop; each thread handles one diagonal element
    if (i < width) {
        tgtMat[width * i + i] = mat[width * i + i] + vec[i];
    }
}