#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kCeil(float* mat, float* target, unsigned int len) {
    // Calculate the global index for each thread
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure each thread handles multiple elements if necessary
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        // Perform ceil operation
        target[i] = ceilf(mat[i]);
    }
}