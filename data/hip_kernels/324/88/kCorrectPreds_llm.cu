#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kCorrectPreds(float* mat, float* p, float* target, unsigned int len, float cutoff) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride-based loop for coalesced memory access
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        float pi = p[i];
        float mati = mat[i];
        target[i] = mati * (pi >= cutoff) + (1 - mati) * (pi < cutoff);
    }
}