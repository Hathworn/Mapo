#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int row = idx / width; // Calculate row index
    const unsigned int col = idx % width; // Calculate column index
    if(row < height && col < width) { // Ensure within bounds
        tgtMat[idx] = mat[idx] * vec[row]; // Multiply with corresponding vector element
    }
}