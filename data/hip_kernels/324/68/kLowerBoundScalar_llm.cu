#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLowerBoundScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop stride using total number of threads
    for (unsigned int i = idx; i < len; i += gridDim.x * blockDim.x) {
        float m_val = mat[i];
        target[i] = m_val < val ? val : m_val;  // Load once, reduce potential latency
    }
}