#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSqrt(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    // Use stride loop to leverage full GPU parallelism
    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = sqrtf(mat[i]); // Use sqrtf for float precision
    }
}