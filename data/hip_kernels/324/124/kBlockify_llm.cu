#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kBlockify(float* source, float* target, int numdims, int blocksize) {
    // Calculate the global thread ID
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = gridDim.x * blockDim.x; // Calculate stride for iteration

    // Iterate over data with a stride to ensure all elements are processed
    for (unsigned int target_ind = idx; target_ind < numdims; target_ind += stride) {
        const int block = target_ind / blocksize;
        target[blockIdx.x * numdims + target_ind] = source[blockIdx.x * numdims + block * blocksize];
    }
}