#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplyCos(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Optimize loop by stride processing each thread's lane efficiently
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = __cosf(mat[i]);
    }
}