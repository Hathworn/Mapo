#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int Nthreads = 1024, maxFR = 5000, NrankMax = 6;

__global__ void cleanup_spikes(const double *Params, const float *err, const int *ftype, float *x, int *st, int *id, int *counter) {
    int lockout = static_cast<int>(Params[4]) - 1;
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    int NT = static_cast<int>(Params[0]);
    int tid0 = bid * blockDim.x;
    float Th = static_cast<float>(Params[2]);

    __shared__ float sdata[Nthreads + 2 * 81 + 1];

    while(tid0 < NT - Nthreads - lockout + 1) {
        // Load data to shared memory, reduce redundant computations
        if (tid < 2 * lockout) {
            sdata[tid] = err[tid0 + tid];
        }
        if (tid0 + tid + 2 * lockout < NT) {
            sdata[tid + 2 * lockout] = err[2 * lockout + tid0 + tid];
        } else {
            sdata[tid + 2 * lockout] = 0.0f;
        }

        __syncthreads();

        int t0 = tid + lockout + tid0;
        float err0 = sdata[tid + lockout];

        if (err0 > Th * Th && t0 < NT - lockout - 1) {
            bool flag = false;

            // Check for a valid spike
            for (int j = -lockout; j <= lockout; ++j) {
                if (sdata[tid + lockout + j] > err0) {
                    flag = true;
                    break;
                }
            }

            if (!flag) {
                int indx = atomicAdd(&counter[0], 1);

                // Store results safely within the bounds of maxFR
                if (indx < maxFR) {
                    st[indx] = t0;
                    id[indx] = ftype[t0];
                    x[indx] = err0;
                }
            }
        }

        // Update starting index for next iteration
        tid0 += blockDim.x * gridDim.x;
    }
}