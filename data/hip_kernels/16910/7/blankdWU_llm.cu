#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024,  NrankMax = 3, nt0max = 71, NchanMax = 1024;

__global__ void blankdWU(const double *Params, const double *dWU, const int *iC, const int *iW, double *dWUblank) {
    // Variable declarations
    int nt0 = (int) Params[4];
    int Nchan = (int) Params[9];
    int NchanNear = (int) Params[10];
    int tidx = threadIdx.x;
    int bid = blockIdx.x;

    // Use shared memory for intermediate calculations (if needed)
    __shared__ int shared_iC[NchanMax];

    if (threadIdx.y < NchanNear) {
        shared_iC[threadIdx.y] = iC[threadIdx.y + NchanNear * iW[bid]];
    }
    __syncthreads();

    for (int tidy = threadIdx.y; tidy < NchanNear; tidy += blockDim.y) {
        int iChan = shared_iC[tidy];
        dWUblank[tidx + nt0 * iChan + bid * nt0 * Nchan] = 
            dWU[tidx + nt0 * iChan + bid * nt0 * Nchan];
    }
}