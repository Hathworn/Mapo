#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int Nthreads = 1024, maxFR = 5000, NrankMax = 6;

__global__ void cleanup_heights(const double *Params, const float *x, const int *st, const int *id, int *st1, int *id1, int *counter) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int Nmax = min(maxFR, counter[0]);
    volatile __shared__ float s_id[maxFR], s_x[maxFR];
    __shared__ int local_counter;
    
    // Load data into shared memory efficiently
    for (int i = tid; i < Nmax; i += blockDim.x) {
        s_x[i] = x[i];
        s_id[i] = id[i];
    }
    __syncthreads();

    // Initialize local counter in shared memory
    if (tid == 0) local_counter = 0;
    __syncthreads();

    if (bid * blockDim.x + tid < Nmax) {
        float xmax = s_x[tid];
        bool flag = true;
        // Optimize loop and use early exit on condition
        for (int t = 0; t < Nmax; t++) {
            int d = abs(s_id[t] - s_id[tid]);
            if (d < 5 && xmax < s_x[t]) {
                flag = false;
                break; // Exit early if condition is met
            }
        }
        // If flag is set, it's the local maximum across nearby channels
        if (flag) {
            int indx = atomicAdd(&local_counter, 1); // Use local_counter for atomic operation
            if (indx < maxFR) {
                st1[indx] = st[tid];
                id1[indx] = s_id[tid];
            }
        }
    }
    __syncthreads();

    // Add local counters to global counter
    if (tid == 0) {
        atomicAdd(&counter[1], local_counter);
    }
}