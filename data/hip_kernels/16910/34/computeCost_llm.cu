#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void computeCost(const double *Params, const float *Ws, const float *mus, const float *W, const float *mu, const bool *iMatch, const int *iC, const int *Wh, float *cmax) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int Nspikes = (int) Params[0];
    int Nchan = (int) Params[7];
    int NrankPC = (int) Params[1];
    int NchanNear = (int) Params[6];
    int Nthreads = blockDim.x;

    while (tid < Nspikes) {
        int my_chan = Wh[tid];
        if (iMatch[my_chan + bid * Nchan]) {
            float xsum = 0.0f;

            // Use registers for frequently accessed variables.
            int base_idx_Ws = NrankPC * NchanNear * tid;
            int base_idx_W = NrankPC * Nchan * bid;

            for (int k = 0; k < NchanNear; ++k) {
                int this_chan = iC[k + NchanNear * my_chan];

                for (int j = 0; j < NrankPC; ++j) {
                    // Minimize repeated index calculations inside the loop.
                    int idx_Ws = j + NrankPC * k + base_idx_Ws;
                    int idx_W = j + NrankPC * this_chan + base_idx_W;
                    xsum += Ws[idx_Ws] * W[idx_W];
                }
            }

            float Ci = mu[bid] * mu[bid] + mus[tid] * mus[tid] - 2 * mus[tid] * mu[bid] * xsum;
            cmax[tid + bid * Nspikes] = Ci;
        }
        tid += Nthreads;
    }
}