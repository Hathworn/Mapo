#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 100000, NrankMax = 3, nmaxiter = 500, NchanMax = 32;

__global__ void average_snips(const double *Params, const int *st, const int *id,  const float *x, const float *y,  const int *counter, const float *dataraw, const float *W, const float *U, double *WU, int *nsp, const float *mu, const float *z) {

    int nt0 = (int) Params[4];
    int Nchan = (int) Params[9];
    int Nrank = (int) Params[6];
    int Nfilt = (int) Params[1];
    int NT = (int) Params[0];
    int tidx = threadIdx.x;
    int bid = blockIdx.x;
    float Th = (float) Params[15];

    for (int currInd = 0; currInd < counter[0]; currInd++) {
        if (x[currInd] > Th && id[currInd] == bid) { // Combined conditions to reduce nesting
            if (tidx == 0 && threadIdx.y == 0) {
                atomicAdd(&nsp[bid], 1); // Use atomic to prevent race condition
            }

            for (int tidy = threadIdx.y; tidy < Nchan; tidy += blockDim.y) { // Use for-loop for channel iteration
                double X = 0.0;
                for (int k = 0; k < Nrank; k++) { // Loop over rank
                    X += W[tidx + bid * nt0 + nt0 * Nfilt * k] * U[tidy + bid * Nchan + Nchan * Nfilt * k];
                }

                double xsum = dataraw[st[currInd] + tidx + NT * tidy] + y[currInd] * X;
                atomicAdd(&WU[tidx + tidy * nt0 + nt0 * Nchan * bid], xsum); // Use atomic for WU update
            }
        }
    }
}