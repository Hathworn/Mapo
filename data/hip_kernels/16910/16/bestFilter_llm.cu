#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 100000, NrankMax = 3, nmaxiter = 500, NchanMax = 32;

__global__ void bestFilter(const double *Params, const float *data, const float *mu, float *err, float *eloss, int *ftype){
    // Use shared memory for frequently accessed variables
    __shared__ float shared_mu[NchanMax];
    
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int NT = (int) Params[0];
    int Nfilt = (int) Params[1];
    float lam = (float) Params[7];
    int nt0 = (int) Params[4];
    
    if (tid < NchanMax) {
        shared_mu[tid] = mu[tid];
    }
    __syncthreads();

    int tid0 = tid + bid * blockDim.x;
    while (tid0 < NT - nt0) {
        float Cbest = 0.0f;
        float Cnextbest = 0.0f;
        int ibest = 0;

        #pragma unroll  // Unroll loop for efficiency
        for (int i = 0; i < Nfilt; i++) {
            float a = 1 + lam;
            float b = fmaxf(0.0f, data[tid0 + NT * i]) + lam * shared_mu[i];
            float Cf = b * b / a - lam * shared_mu[i] * shared_mu[i];

            if (Cf > Cbest + 1e-6f) {
                Cnextbest = Cbest;
                Cbest = Cf;
                ibest = i;
            }
            else if (Cf > Cnextbest + 1e-6f) {
                Cnextbest = Cf;
            }
        }
        err[tid0] = Cbest;
        eloss[tid0] = Cbest - Cnextbest;
        ftype[tid0] = ibest;

        tid0 += blockDim.x * gridDim.x;
    }
}