#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int Nthreads = 1024, maxFR = 5000, NrankMax = 6;

__global__ void sumChannels(const double *Params, const float *data, float *datasum, int *kkmax, const int *iC) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int NT = (int) Params[0];
    int Nchan = (int) Params[9];
    int Nsum = (int) Params[13];
    int Nrank = (int) Params[14];
    int NchanNear = (int) Params[10];

    // Iterate over all threads in strides of total threads
    for (int tid0 = tid; tid0 < NT; tid0 += blockDim.x * gridDim.x) {
        for (int i = 0; i < Nchan; i++) {
            float Cmax = 0.0f;
            int kmax = 0;
            
            // Loop unrolling for better performance
            for (int t = 0; t < Nrank; t++) {
                float Cf = 0.0f;
                for (int j = 0; j < Nsum; j++) {
                    int iChan = iC[j + NchanNear * i];
                    Cf += data[tid0 + NT * iChan + t * NT * Nchan];
                    float CfNorm = Cf * Cf / (1 + j); // Pre-calculate for reuse
                    if (CfNorm > Cmax) {
                        Cmax = CfNorm;
                        kmax = j + t * Nsum;
                    }
                }
            }
            datasum[tid0 + NT * i] = Cmax;
            kkmax[tid0 + NT * i] = kmax;
        }
    }
}