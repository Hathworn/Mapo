#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int Nthreads = 1024, maxFR = 5000, NrankMax = 6;

__global__ void Conv1D(const double *Params, const float *data, const float *W, float *conv_sig) {
    extern __shared__ float shared_mem[]; // Combined shared memory
    float* sW = shared_mem; // Shared memory for W
    float* sdata = sW + 81 * NrankMax; // Shared memory for data after W
    float y;
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int NT = (int) Params[0];
    int Nrank = (int) Params[14];
    int nt0 = (int) Params[4];
    int Nchan = (int) Params[9];

    // Load W into shared memory in unrolled loops
    if (tid < nt0 * Nrank) {
        sW[tid] = W[tid];
    }
    __syncthreads();

    int tid0 = 0;
    while (tid0 < NT - Nthreads - nt0 + 1) {
        if (tid < nt0) {
            sdata[tid] = data[tid0 + tid + NT * bid];
        }
        sdata[tid + nt0] = data[nt0 + tid0 + tid + NT * bid];
        __syncthreads();

        for (int nid = 0; nid < Nrank; nid++) {
            y = 0.0f;
            #pragma unroll 4
            for (int i = 0; i < nt0; i++) {
                y += sW[i + nid * nt0] * sdata[i + tid];
            }
            // Write result back to global memory
            conv_sig[tid0 + tid + NT * bid + nid * NT * Nchan] = y;
        }
        tid0 += Nthreads;
        __syncthreads();
    }
}