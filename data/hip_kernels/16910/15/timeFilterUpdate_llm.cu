#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 100000, NrankMax = 3, nmaxiter = 500, NchanMax = 32;

__global__ void timeFilterUpdate(const double *Params, const float *data, const float *W, const bool *UtU, float *conv_sig, const int *st, const int *id, const int *counter){

    volatile __shared__ float  sW[81*NrankMax];
    float x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int NT = (int)Params[0];
    int Nfilt = (int)Params[1];
    int Nrank = (int)Params[6];
    int nt0 = (int)Params[4];

    // Load W into shared memory
    if (tid < nt0) {
        for (int k = 0; k < Nrank; k++) {
            sW[tid + k * nt0] = W[tid + nt0 * bid + nt0 * Nfilt * k];
        }
    }
    __syncthreads();

    // Optimized loop with shared memory prefetching
    for (int ind = counter[1]; ind < counter[0]; ind++) {
        if (UtU[id[ind] + Nfilt * bid]) {
            int tid0 = st[ind] - nt0 + tid;

            if (tid0 >= 0 && tid0 < NT - nt0) {
                x = 0.0f;

                for (int k = 0; k < Nrank; k++) {
                    #pragma unroll // Unroll loop for better performance
                    for (int t = 0; t < nt0; t++) {
                        x += sW[t + k * nt0] * data[t + tid0 + NT * bid + NT * Nfilt * k];
                    }
                }

                conv_sig[tid0 + NT * bid] = x;
            }
        }
    }
}