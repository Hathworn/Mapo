#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 100000, NrankMax = 3, nmaxiter = 500, NchanMax = 32;

__global__ void spaceFilterUpdate(const double *Params, const float *data, const float *U, const bool *UtU, const int *iC, const int *iW, float *dprod, const int *st, const int *id, const int *counter) {
    extern __shared__ float sU[];
    __shared__ int iU[NchanMax];
    float x;
    int tid, bid, ind, t, k, Nrank, NT, Nfilt, NchanU, Nchan;

    tid = threadIdx.x;
    bid = blockIdx.x;
    NT = static_cast<int>(Params[0]);
    Nfilt = static_cast<int>(Params[1]);
    Nrank = static_cast<int>(Params[6]);
    NchanU = static_cast<int>(Params[10]);
    int nt0 = static_cast<int>(Params[4]);
    Nchan = static_cast<int>(Params[9]);

    // Preload iC and U for current batch
    if (tid < NchanU) {
        iU[tid] = iC[tid + NchanU * iW[bid]];
        for (k = 0; k < Nrank; k++) {
            sU[tid + k * NchanU] = U[iU[tid] + Nchan * bid + Nchan * Nfilt * k];
        }
    }
    __syncthreads();

    for (ind = counter[1]; ind < counter[0]; ind++) {
        if (UtU[id[ind] + Nfilt * bid]) {
            t = st[ind] + tid - nt0;
            // Directly checking boundary within the loop
            if (t >= 0 && t < NT) {
                for (k = 0; k < Nrank; k++) {
                    x = 0.0f;
                    for (int i = 0; i < NchanU; i++) {
                        x += sU[i + NchanU * k] * data[t + NT * iU[i]];
                    }
                    dprod[t + NT * bid + k * NT * Nfilt] = x;
                }
            }
        }
    }
}