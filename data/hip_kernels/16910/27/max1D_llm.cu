#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 10000, NrankMax = 3, nt0max=81, NchanMax = 17;

__global__ void max1D(const double *Params, const float *data, float *conv_sig) {

    extern __shared__ float sdata[];
    float y, spkTh;
    int tid, tid0, bid, i, NT, nt0;

    NT      = (int) Params[0];
    nt0     = (int) Params[3];
    spkTh   = (float) Params[5];
    tid     = threadIdx.x;
    bid     = blockIdx.x;

    tid0 = bid * Nthreads; // Adjust threads to cover multiple blocks
    while (tid0 < NT - nt0 + 1) {
        // Load data into shared memory efficiently
        if (tid < nt0)
            sdata[tid] = data[tid0 + tid];
        if (tid + nt0 < Nthreads + nt0)
            sdata[tid + nt0] = data[tid0 + tid + nt0];
        __syncthreads();

        y = 0.0f;
        #pragma unroll 4
        for (i = 0; i < nt0; i++)
            y = max(y, sdata[tid + i]);

        // Only write back if condition is met
        if (y > spkTh && tid0 + tid < NT)
            conv_sig[tid0 + tid] = y;

        tid0 += Nthreads * gridDim.x; // Move to next block of data
        __syncthreads();
    }
}