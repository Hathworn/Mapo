#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void count_spikes(const double *Params, const int *id, int *nsp, const float *x, float *V) {
    int tid = threadIdx.x; // Thread index
    int bid = blockIdx.x;  // Block index
    int NthreadsMe = blockDim.x; // Threads per block
    int Nblocks = gridDim.x; // Total blocks

    int Nspikes = (int) Params[0];
    int Nfilters = (int) Params[2];

    int tind = tid + NthreadsMe * bid;

    // Loop stride by total number of threads
    while (tind < Nfilters) {
        nsp[tind] = 0; // Initialize spike count
        V[tind] = 0.0f; // Initialize voltage sum
        for (int ind = 0; ind < Nspikes; ind++) {
            if (id[ind] == tind) {
                nsp[tind]++;
                V[tind] += x[ind]; // Accumulate voltage for spikes relating to this filter
            }
        }
        V[tind] = V[tind] / (.001f + (float) nsp[tind]); // Compute average voltage
        tind += NthreadsMe * Nblocks;
    }
}