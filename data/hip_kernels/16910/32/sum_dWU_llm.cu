#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_dWU(const double *Params, const float *bigArray, float *WU) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    
    int Nfeatures = (int) Params[1];
    int NfeatW = (int) Params[4];
    int Nfilters = (int) Params[2];
    int Nthreads = blockDim.x;
    int Nblocks = gridDim.x;

    int nWU = NfeatW * Nfilters;
    int nElem = Nfeatures * NfeatW;

    int tind = tid + bid * Nthreads;

    while (tind < nWU) {
        int currFilt = tind / NfeatW; // More efficient integer division
        int currFW = tind % NfeatW;   // More efficient modulo operation

        float sum = 0.0f;

        // Using a strided loop for better performance
        for (int ind = 0; ind < Nfeatures; ++ind) {
            int currIndex = currFilt * nElem + ind * NfeatW + currFW;
            sum += bigArray[currIndex];
        }

        WU[tind] += sum;
        tind += Nblocks * Nthreads;
    }
}