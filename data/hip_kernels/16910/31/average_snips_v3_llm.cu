#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void average_snips_v3(const double *Params, const int *ioff, const int *id, const float *uproj, const float *cmax, float *bigArray) {
    // Get thread and block indices for feature and filter
    int tid = threadIdx.x; // Feature index
    int bid = blockIdx.x;  // Filter index

    // Load parameters
    int Nspikes = static_cast<int>(Params[0]);
    int Nfeatures = static_cast<int>(Params[1]);
    int NfeatW = static_cast<int>(Params[4]);

    // Pointer to the section of bigArray for a specific filter and feature
    float* bigArrayPointer = bigArray + tid * (NfeatW + 1) + Nfeatures * NfeatW * bid;

    // Loop over spikes and compute projection sum
    for (int ind = 0; ind < Nspikes; ind++) {
        if (id[ind] == bid) {
            float xsum = uproj[tid + Nfeatures * ind];
            atomicAdd(&bigArrayPointer[ioff[ind]], xsum); // Use atomic add for thread safety
        }
    }
}