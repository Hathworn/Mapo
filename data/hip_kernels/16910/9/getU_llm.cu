#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int Nthreads = 1024, NrankMax = 3, nt0max = 71, NchanMax = 1024;

__global__ void getU(const double *Params, const double *dWU, double *W, double *U) {

    int Nfilt, nt0, tidx, tidy, bid, Nchan, k;
    double x;

    nt0 = (int) Params[4];
    Nchan = (int) Params[9];
    Nfilt = (int) Params[1];
    tidx = threadIdx.x;
    tidy = threadIdx.y + blockIdx.y * blockDim.y;  // Use 2D grid for more threads
    bid = blockIdx.x;

    if (tidy < Nchan) {  // Avoid out-of-bounds access
        x = 0.0f;
        for (k = 0; k < nt0; k++) {
            x += W[k + nt0 * bid + nt0 * Nfilt * tidx] *
                 dWU[k + tidy * nt0 + bid * Nchan * nt0];
        }
        U[tidy + Nchan * bid + Nchan * Nfilt * tidx] = x;
    }
}