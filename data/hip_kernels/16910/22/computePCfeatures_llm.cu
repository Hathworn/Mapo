#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 100000, NrankMax = 3, nmaxiter = 500, NchanMax = 32;
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////

// THIS UPDATE DOES NOT UPDATE ELOSS?
//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////






//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void	computePCfeatures(const double *Params, const int *counter, const float *dataraw,  const int *st, const int *id, const float *x, const float *W, const float *U, const float *mu, const int *iW, const int *iC, const float *wPCA, float *featPC){

    // Using shared memory effectively
    __shared__ float sPCA[81 * NrankMax], sW[81 * NrankMax], sU[NchanMax * NrankMax];
    __shared__ int iU[NchanMax];

    int bid, nt0, t, tidx, tidy, k, NT, ind, Nchan, NchanU, Nfilt, Nrank;
    float X = 0.0f, Y = 0.0f;

    NT = (int)Params[0];
    nt0 = (int)Params[4];
    Nchan = (int)Params[9];
    Nfilt = (int)Params[1];
    Nrank = (int)Params[6];
    NchanU = (int)Params[10];

    tidx = threadIdx.x;
    tidy = threadIdx.y;
    bid = blockIdx.x;

    if (tidy == 0)
        iU[tidx] = iC[tidx + NchanU * iW[bid]];
    __syncthreads();

    int iUOffset = tidx + tidy * NchanU;
    sU[iUOffset] = U[iU[tidx] + Nchan * bid + Nchan * Nfilt * tidy];

    for (int i = tidx; i < nt0; i += blockDim.x) {
        int offset = i + tidy * nt0;
        sW[offset] = W[i + bid * nt0 + Nfilt * nt0 * tidy];
        sPCA[offset] = wPCA[i + nt0 * tidy];
    }

    __syncthreads();

    // Compute wPCA projections of the filter
    Y = 0.0f;
    for (k = 0; k < Nrank; k++) {
        X = 0.0f;
        for (t = 0; t < nt0; t++)
            X += sW[t + k * nt0] * sPCA[t + tidy * nt0];
        Y += X * sU[tidx + k * NchanU];
    }

    // Compute features for each matching spike
    for (ind = 0; ind < counter[0]; ind++) {
        if (id[ind] == bid) {
            X = Y * x[ind];
            for (t = 0; t < nt0; t++)
                X += dataraw[st[ind] + t + NT * iU[tidx]] * sPCA[t + nt0 * tidy];
            featPC[tidx + tidy * NchanU + ind * NchanU * Nrank] = X;
        }
    }
}