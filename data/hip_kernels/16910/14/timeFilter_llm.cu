#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 100000, NrankMax = 3, nmaxiter = 500, NchanMax = 32;

__global__ void	timeFilter(const double *Params, const float *data, const float *W, float *conv_sig) {
    extern __shared__ float shared_mem[];
    float *sW = shared_mem;
    float *sdata = &sW[81 * NrankMax];
    float x;
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int NT = (int) Params[0];
    int Nfilt = (int) Params[1];
    int Nrank = (int) Params[6];
    int nt0 = (int) Params[4];
    int tid0;

    // Load weights into shared memory
    if (tid < nt0 * Nrank) {
        sW[tid] = W[tid % nt0 + (bid + Nfilt * (tid / nt0)) * nt0];
    }

    __syncthreads();

    for (tid0 = 0; tid0 < NT - Nthreads - nt0 + 1; tid0 += Nthreads) {
        // Load data into shared memory
        if (tid < nt0 * NrankMax) {
            sdata[tid % nt0 + (tid / nt0) * (Nthreads + nt0)] = data[tid0 + tid % nt0 + NT * (bid + Nfilt * (tid / nt0))];
        }
        #pragma unroll 3
        for (int nid = 0; nid < Nrank; nid++) {
            sdata[tid + nt0 + nid * (Nthreads + nt0)] = data[nt0 + tid0 + tid + NT * (bid + nid * Nfilt)];
        }
        __syncthreads();

        x = 0.0f;
        #pragma unroll 4
        for (int nid = 0; nid < Nrank; nid++) {
            #pragma unroll 4
            for (int i = 0; i < nt0; i++) {
                x += sW[i + nid * nt0] * sdata[i + tid + nid * (Nthreads + nt0)];
            }
        }

        conv_sig[tid0 + tid + NT * bid] = x;
        __syncthreads();
    }
}