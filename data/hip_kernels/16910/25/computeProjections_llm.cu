#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int Nthreads = 1024, maxFR = 10000, NrankMax = 3, nt0max = 81, NchanMax = 17;

__global__ void computeProjections(const double *Params, const float *dataraw, const int *iC, const int *st, const int *id, const float *W, float *feat) {
    float x;
    int tidx, tidy, my_chan, this_chan, tid, bid, nt0, NchanNear, j, NT, NrankPC;
    __shared__ float sW[nt0max * NrankMax];  // Allocate shared memory for W
    __shared__ float sD[nt0max * NchanMax];  // Allocate shared memory for data

    NT = (int) Params[0];
    NchanNear = (int) Params[2];
    nt0 = (int) Params[3];
    NrankPC = (int) Params[6];

    tidx = threadIdx.x;
    tidy = threadIdx.y;
    bid = blockIdx.x;

    // Move wPCA to shared memory
    if (tidx < nt0) {
        sW[tidx + tidy * nt0] = W[tidx + tidy * nt0];
    }
    __syncthreads();  // Synchronize threads after loading W

    tid = tidx + tidy * blockDim.x;

    // Move raw data to shared memory
    while (tid < nt0) {
        my_chan = id[bid];
        for (j = 0; j < NchanNear; j++) {
            this_chan = iC[j + NchanNear * my_chan];
            sD[tid + nt0 * j] = dataraw[tid + st[bid] + Params[4] - 1 + NT * this_chan];
        }
        tid += blockDim.x * blockDim.y;
    }
    __syncthreads();  // Synchronize threads after loading data

    x = 0.0f;
    for (int t = 0; t < nt0; t++) {
        x += sD[t + nt0 * tidx] * sW[t + nt0 * tidy];
    }

    // Store the results
    if (tidy < NrankPC && tidx < NchanNear) {
        feat[tidy + tidx * NrankPC + NrankPC * NchanNear * bid] = x;
    }
}