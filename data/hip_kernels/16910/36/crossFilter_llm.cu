#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int nblock = 32;
//////////////////////////////////////////////////////////////////////////////////////////

__global__ void crossFilter(const double *Params, const float *W1, const float *W2, const float *UtU, float *WtW) {
    __shared__ float shW1[nblock * 81], shW2[nblock * 81];

    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int bidx = blockIdx.x;
    int bidy = blockIdx.y;

    int Nfilt = static_cast<int>(Params[1]);
    int nt0 = static_cast<int>(Params[9]);

    int tid1 = tidx + bidx * nblock;
    int tid2 = tidy + bidy * nblock;

    if (tid2 < Nfilt) {
        for (int loadIdx = tidx; loadIdx < nt0; loadIdx += blockDim.x) {
            shW1[loadIdx + tidy * nt0] = W1[loadIdx + tid2 * nt0];
            shW2[loadIdx + tidy * nt0] = W2[loadIdx + tid2 * nt0];
        }
    }

    __syncthreads();

    if (tid2 < Nfilt && tid1 < Nfilt) {
        for (int i = 0; i < 2 * nt0 - 1; i++) {
            float x = 0.0f;
            int max_t = min(nt0, i + 1);
            int start_t = max(0, i - nt0 + 1);
            for (int t = start_t; t < max_t; t++) {
                x += shW1[t + nt0 * tidx] * shW2[t + (nt0 - i - 1) + nt0 * tidy];
            }
            WtW[tid1 + tid2 * Nfilt + i * Nfilt * Nfilt] = x * UtU[tid1 + tid2 * Nfilt];
        }
    }
}