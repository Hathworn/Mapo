#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 100000, NrankMax = 3, nmaxiter = 500, NchanMax = 32;

__global__ void bestFilterUpdate(const double *Params, const float *data, const float *mu, float *err, float *eloss, int *ftype, const int *st, const int *id, const int *counter) {
    int tid = threadIdx.x;
    int NT = (int) Params[0];
    int Nfilt = (int) Params[1];
    float lam = (float) Params[7];
    int nt0 = (int) Params[4];

    int ind = counter[1] + blockIdx.x;

    if (ind < counter[0]) {
        int t = st[ind] - nt0 + tid;
        if (t >= 0 && t < NT) {
            float Cbest = -1.0f;  // Improves comparison and ensures updates
            float Cnextbest = -1.0f;  // Tracks second best score
            int ibest = -1;  // Tracks index of the best score

            for (int i = 0; i < Nfilt; i++) {
                float a = 1 + lam;
                float b = max(0.0f, data[t + NT * i]) + lam * mu[i];

                float Cf = b * b / a - lam * mu[i] * mu[i];

                if (Cf > Cbest) {
                    Cnextbest = Cbest;  // Update second best
                    Cbest = Cf;  // Update best
                    ibest = i;  // Update index
                } else if (Cf > Cnextbest) {
                    Cnextbest = Cf;  // Update second best only
                }
            }
            err[t] = Cbest;
            ftype[t] = ibest;
        }
    }
}