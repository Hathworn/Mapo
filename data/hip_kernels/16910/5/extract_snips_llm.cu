#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int Nthreads = 1024, maxFR = 5000, NrankMax = 6;

__global__ void extract_snips(const double *Params, const int *st, const int *id, const int *counter, const float *dataraw, float *WU) {

    int nt0, tidx, bid, ind, NT, Nchan, Nmax;
    NT = (int) Params[0];
    nt0 = (int) Params[4];
    Nchan = (int) Params[9];

    tidx = threadIdx.x;
    bid = blockIdx.x;

    Nmax = min(maxFR, counter[1]);

    // Use shared memory for better memory access patterns
    extern __shared__ float shared_data[];
    
    for (ind = 0; ind < Nmax; ind++) {
        if (id[ind] == bid) {
            int offset = nt0 * Nchan * ind;
            for (int ch = threadIdx.y; ch < Nchan; ch += blockDim.y) {
                // Load data into shared memory for coalesced access
                shared_data[tidx + ch * blockDim.x] = dataraw[st[ind] + tidx + NT * ch];
                
                // Synchronize threads to ensure shared memory is fully populated
                __syncthreads();

                // Write from shared memory to global memory with coalesced writes
                WU[tidx + ch * nt0 + offset] = shared_data[tidx + ch * blockDim.x];
                
                // Synchronize threads before next operation
                __syncthreads();
            }
        }
    }
}