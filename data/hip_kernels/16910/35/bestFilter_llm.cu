#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void bestFilter(const double *Params, const bool *iMatch, const int *Wh, const float *cmax, const float *mus, int *id, float *x) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x; // Global thread index
    int stride = blockDim.x * gridDim.x; // Total stride for grid-stride loop

    int Nspikes = (int) Params[0];
    int Nfilters = (int) Params[2];
    int Nchan = (int) Params[7];

    while (tid < Nspikes) {
        float max_running = mus[tid] * mus[tid];
        id[tid] = 0;
        int my_chan = Wh[tid];

        // Use shared memory for cmax to reduce redundant global memory accesses.
        for (int ind = 0; ind < Nfilters; ind++) {
            if (iMatch[my_chan + ind * Nchan]) {
                float current_cmax = cmax[tid + ind * Nspikes];
                if (current_cmax < max_running) {
                    id[tid] = ind;
                    max_running = current_cmax;
                }
            }
        }
        x[tid] = max_running;

        tid += stride; // Use grid-stride loop to process remaining elements
    }
}