#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int Nthreads = 1024, maxFR = 100000, NrankMax = 3, nmaxiter = 500, NchanMax = 32;

__global__ void subtract_spikes(const double *Params, const int *st, const int *id, const float *x, const int *counter, float *dataraw, const float *W, const float *U) {
    int nt0, tidx, tidy, k, NT, ind, Nchan, Nfilt, Nrank;
    float X;

    NT    = (int) Params[0];
    nt0   = (int) Params[4];
    Nchan = (int) Params[9];
    Nfilt = (int) Params[1];
    Nrank = (int) Params[6];

    tidx = threadIdx.x;
    ind  = counter[1] + blockIdx.x;

    // Optimize loop with stride approach for better parallel execution
    for (; ind < counter[0]; ind += gridDim.x) {
        for (tidy = threadIdx.y; tidy < Nchan; tidy += blockDim.y) {
            X = 0.0f;
            #pragma unroll // Unroll loop for better performance
            for (k = 0; k < Nrank; k++) {
                X += W[tidx + id[ind] * nt0 + nt0 * Nfilt * k] * U[tidy + id[ind] * Nchan + Nchan * Nfilt * k];
            }
            dataraw[tidx + st[ind] + NT * tidy] -= x[ind] * X;
        }
    }
}