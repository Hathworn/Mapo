#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeCost(const double *Params, const float *uproj, const float *mu, const float *W, const bool *match, const int *iC, const int *call, float *cmax){

    int tid = threadIdx.x + blockIdx.x * blockDim.x; // Flatten indexing
    int stride = gridDim.x * blockDim.x; // Stride for grid-stride loop

    int Nspikes = (int) Params[0];
    int NrankPC = (int) Params[1];
    float lam = (float) Params[5];
    int NchanNear = (int) Params[6];
    int Nchan = (int) Params[7];

    while(tid < Nspikes) {
        int my_chan = call[tid];
        if (match[my_chan + blockIdx.x * Nchan]) {
            float xsum = 0.0f;
            for (int k = 0; k < NchanNear; k++) {
                int this_chan = iC[k + my_chan * NchanNear];
                for (int j = 0; j < NrankPC; j++) {
                    xsum += uproj[j + NrankPC * k + NrankPC * NchanNear * tid] *
                            W[j + NrankPC * this_chan + NrankPC * Nchan * blockIdx.x];
                }
            }
            float Ci = max(0.0f, xsum) + lam / mu[blockIdx.x];
            cmax[tid + blockIdx.x * Nspikes] = Ci * Ci / (1.0f + lam/(mu[blockIdx.x] * mu[blockIdx.x])) - lam;
        }
        tid += stride;
    }
}