#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int Nthreads = 1024, maxFR = 100000, NrankMax = 3, nmaxiter = 500, NchanMax = 32;

__global__ void cleanup_spikes(const double *Params, const float *data, const float *mu, const float *err, const float *eloss, const int *ftype, int *st, int *id, float *x, float *y, float *z, int *counter) {

    int lockout, indx, tid, NT, tid0, j, id0, t0;
    volatile __shared__ float sdata[Nthreads + 2 * 81 + 1];
    bool flag = false;
    float err0, Th;

    lockout = (int) Params[4] - 1;
    tid = threadIdx.x;

    // Pre-calculate block index * block size to avoid repetitive multiplication
    int blockSize = blockDim.x;
    int blockStart = blockIdx.x * blockSize;

    NT = (int) Params[0];
    Th = (float) Params[2];

    // Loop over data chunks
    while (blockStart < NT - Nthreads - lockout + 1) {
        tid0 = blockStart;

        // Load data into shared memory in coalesced manner
        if (tid < 2 * lockout)
            sdata[tid] = err[tid0 + tid];
        sdata[tid + 2 * lockout] = err[2 * lockout + tid0 + tid];

        __syncthreads();

        err0 = sdata[tid + lockout];
        if (err0 > Th * Th) {
            flag = false;

            // Use parallel reduction technique for maximum comparison
            #pragma unroll
            for (j = -lockout; j <= lockout; j++) {
                if (sdata[tid + lockout + j] > err0) {
                    flag = true;
                    break;
                }
            }

            if (!flag) {
                indx = atomicAdd(&counter[0], 1);
                if (indx < maxFR) {
                    t0 = tid + lockout + tid0;
                    id0 = ftype[t0];
                    st[indx] = t0;
                    id[indx] = id0;
                    y[indx] = data[t0 + NT * id0];

                    x[indx] = sqrt(err0);
                    z[indx] = eloss[t0];
                }
            }
        }

        // Increment the block start index by total blocks size
        blockStart += blockSize * gridDim.x;
    }
}