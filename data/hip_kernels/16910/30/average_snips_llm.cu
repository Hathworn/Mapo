#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void average_snips(const double *Params, const int *iC, const int *call, const int *id, const float *uproj, const float *cmax, float *WU){

    int Nspikes = (int) Params[0];
    int NrankPC = (int) Params[1];
    int NchanNear = (int) Params[6];
    int Nchan = (int) Params[7];

    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int bid = blockIdx.x;

    float xsum = 0.0f;

    // Cache my_chan and this_chan for current block and thread y
    int my_chan = call[bid]; // Access call[bid] once
    int base_index = NchanNear * my_chan; // Precompute base index

    // Loop optimization: exit if id[ind] != bid early
    for (int ind = 0; ind < Nspikes; ++ind) {
        if (id[ind] == bid) {
            int this_chan = iC[tidy + base_index]; // Use cached base_index
            xsum = uproj[tidx + NrankPC * tidy + NrankPC * NchanNear * ind];
            int wu_index = tidx + NrankPC * this_chan + NrankPC * Nchan * bid;
            atomicAdd(&WU[wu_index], xsum); // Use atomic operation for concurrent access
        }
    }
}