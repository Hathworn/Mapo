#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 100000, NrankMax = 3, nmaxiter = 500, NchanMax = 32;

__global__ void spaceFilter(const double *Params, const float *data, const float *U, const int *iC, const int *iW, float *dprod){
    extern __shared__ float sU[]; // Use dynamic shared memory
    __shared__ int iU[32];
    float x;
    int tid, bid, i, k, Nrank, Nchan, NT, Nfilt, NchanU;

    tid = threadIdx.x;
    bid = blockIdx.x;
    NT = (int) Params[0];
    Nfilt = (int) Params[1];
    Nrank = (int) Params[6];
    NchanU = (int) Params[10];
    Nchan = (int) Params[9];

    if (tid < NchanU)
        iU[tid] = iC[tid + NchanU * iW[bid]];
    __syncthreads();

    if(tid < NchanU * Nrank)
        sU[tid] = U[iU[tid % NchanU] + Nchan * bid + Nchan * Nfilt * (tid / NchanU)];
    __syncthreads();

    for (int t = tid; t < NT; t += blockDim.x) { // Unroll the loop for better memory access
        for (k = 0; k < Nrank; k++) {
            x = 0.0f;
            for (i = 0; i < NchanU; i++)
                x += sU[i + NchanU * k] * data[t + NT * iU[i]];
            dprod[t + NT * bid + k * NT * Nfilt] = x;
        }
    }
}