#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 100000, NrankMax = 3, nmaxiter = 500, NchanMax = 32;

__global__ void extractFEAT(const double *Params, const int *st, const int *id, const int *counter, const float *dout, const int *iList, const float *mu, float *d_feat) {
    int tidx = threadIdx.x;
    int tidy = threadIdx.y;
    int bid = blockIdx.x;

    int NT = (int)Params[0];
    int Nnearest = (int)Params[5];
    int NthreadsX = blockDim.x;
    int Nblocks = gridDim.x;
    float lam = (float)Params[7];

    // Calculate initial index based on current block and thread
    int ind = counter[1] + tidx + NthreadsX * bid;

    // Loop through all new spikes
    while (ind < counter[0]) {
        int tcurr = st[ind];
        float rMax = 0.0f;
        int idF = iList[tidy + Nnearest * id[ind]];

        // Minimize shared computation within the loop
        float mu_idF = mu[idF];
        float sqrt_term = sqrtf(lam / (mu_idF * mu_idF) + 1.0f);

        // Loop unrolling for performance improvement
        for (int t = -3; t < 3; t++) {
            float Ci = dout[tcurr + t + idF * NT] + lam / mu_idF;
            float Cf = Ci / sqrt_term;
            rMax = fmaxf(rMax, Cf);
        }
        
        // Write back the computed maximum feature
        d_feat[tidy + ind * Nnearest] = rMax;

        // Move to the next spike index for this thread
        ind += NthreadsX * Nblocks;
    }
}