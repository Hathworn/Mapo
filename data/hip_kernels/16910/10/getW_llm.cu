#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024,  NrankMax = 3, nt0max = 71, NchanMax = 1024;

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void getW(const double *Params, double *wtw, double *W){

    int Nfilt, nt0, tid, bid, Nrank, tmax;
    double x, x0, xmax;
    __shared__ double sW[nt0max * NrankMax], swtw[nt0max * nt0max], xN[1];

    nt0    = (int) Params[4];
    Nrank  = (int) Params[6];
    Nfilt  = (int) Params[1];
    tmax   = (int) Params[11];

    tid = threadIdx.x;
    bid = blockIdx.x;

    // Optimized shared memory loading
    for (int k = tid; k < nt0 * nt0; k += Nthreads)
        swtw[k] = wtw[k + bid * nt0 * nt0];
    
    for (int k = tid; k < Nrank * nt0; k += Nthreads)
        sW[k] = W[k + bid * nt0 * Nfilt];

    __syncthreads();

    // For each SVD rank
    for (int k = 0; k < Nrank; k++) {
        for (int i = 0; i < 100; i++) {
            x = 0.0f;
            // Compute projection of wtw
            for (int t = tid; t < nt0; t += Nthreads) // Partial reduction in thread loop
                x += swtw[tid + t * nt0] * sW[t + k * nt0];

            atomicAdd(&xN[0], x); // Use atomic addition for partial results

            __syncthreads();

            if (i < 99) {
                x0 = 0.00001f;
                if (tid == 0) xN[0] = 0.0f;

                __syncthreads();

                for (int t = tid; t < nt0; t += Nthreads)
                    atomicAdd(&xN[0], sW[t + k * nt0] * sW[t + k * nt0]);

                __syncthreads();

                if (tid == 0) xN[0] = sqrt(xN[0]);

                __syncthreads();

                for (int t = tid; t < nt0; t += Nthreads)
                    sW[t + k * nt0] = xN[0] ? x / xN[0] : 0.0; // Avoid division by zero
            }
            __syncthreads();
        }

        // Subtract off this SVD from wtw
        for (int t = tid; t < nt0; t += Nthreads)
            swtw[tid + t * nt0] -= sW[t + k * nt0] * x;

        __syncthreads();
    }

    xmax = sW[tmax];
    __syncthreads();

    for (int t = tid; t < nt0; t += Nthreads)
        sW[t] = -sW[t] * copysign(1.0, xmax);

    // Write W back
    for (int k = tid; k < Nrank * nt0; k += Nthreads)
        W[k + bid * nt0 * Nfilt] = sW[k];
}