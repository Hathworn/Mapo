#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 5000, NrankMax = 6;

__global__ void  bestFilter(const double *Params, const float *data, float *err, int *ftype, int *kkmax, int *kall) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int NT = (int) Params[0];
    int Nchan = (int) Params[9];

    int tid0 = tid + bid * blockDim.x;

    // Use simple loop unrolling to decrease branching
    for (; tid0 < NT; tid0 += blockDim.x * gridDim.x) {
        float Cbest = 0.0f;
        int ibest = 0;
        int kbest = 0;

        for (int i = 0; i < Nchan; i++) {
            float Cf = data[tid0 + NT*i];
            if (Cf > Cbest + 1e-6) {
                Cbest = Cf;
                ibest = i;
                kbest = kkmax[tid0 + NT*i];
            }
        }
        err[tid0] = Cbest;
        ftype[tid0] = ibest;
        kall[tid0] = kbest;
    }
}