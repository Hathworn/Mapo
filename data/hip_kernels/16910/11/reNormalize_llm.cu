#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024,  NrankMax = 3, nt0max = 71, NchanMax = 1024;

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void reNormalize(const double *Params, const double *A, const double *B, double *W, double *U, double *mu) {

    int Nfilt, nt0, tid, bid, Nchan, k, Nrank, imax, t, tmax;
    double x, xmax, xshift, sgnmax;
    volatile __shared__ double sW[NrankMax * nt0max], sU[NchanMax * NrankMax], sS[NrankMax + 1], sWup[nt0max * 10];

    nt0 = static_cast<int>(Params[4]);
    Nchan = static_cast<int>(Params[9]);
    Nfilt = static_cast<int>(Params[1]);
    Nrank = static_cast<int>(Params[6]);
    tmax = static_cast<int>(Params[11]);
    bid = blockIdx.x;
    tid = threadIdx.x;

    // Load shared memory
    for (k = 0; k < Nrank; k++)
        sW[tid + k * nt0] = W[tid + bid * nt0 + k * Nfilt * nt0];

    while (tid < Nchan * Nrank) {
        sU[tid] = U[tid % Nchan + bid * Nchan + (tid / Nchan) * Nchan * Nfilt];
        tid += blockDim.x;
    }

    __syncthreads();

    // Calculate normalization factor for U
    tid = threadIdx.x;
    if (tid < Nrank) {
        x = 0.0;
        for (k = 0; k < Nchan; k++)
            x += sU[k + tid * Nchan] * sU[k + tid * Nchan];
        sS[tid] = sqrt(x);
    }

    if (tid == 0) {
        x = 0.0000001;
        for (k = 0; k < Nrank; k++)
            x += sS[k] * sS[k];
        sS[Nrank] = sqrt(x);
        mu[bid] = sS[Nrank];
    }

    __syncthreads();

    // Re-normalize U
    tid = threadIdx.x;
    while (tid < Nchan * Nrank) {
        U[tid % Nchan + bid * Nchan + (tid / Nchan) * Nchan * Nfilt] = sU[tid] / sS[Nrank];
        tid += blockDim.x;
    }

    __syncthreads();

    // Find max element and align W
    xmax = 0.0;
    imax = 0;
    for (t = 0; t < nt0; t++)
        if (abs(sW[t]) > xmax) {
            xmax = abs(sW[t]);
            imax = t;
        }

    tid = threadIdx.x;
    for (k = 0; k < Nrank; k++) {
        int ishift = tid + (imax - tmax);
        ishift = (ishift % nt0 + nt0) % nt0;
        xshift = sW[ishift + k * nt0];
        W[tid + bid * nt0 + k * nt0 * Nfilt] = xshift;
    }

    __syncthreads();
    for (k = 0; k < Nrank; k++)
        sW[tid + k * nt0] = W[tid + bid * nt0 + k * nt0 * Nfilt];

    __syncthreads();

    // Compute subsample peak
    tid = threadIdx.x;
    if (tid < 10) {
        sWup[tid] = 0;
        for (t = 0; t < nt0; t++)
            sWup[tid] += A[tid + t * 10] * sW[t];
    }
    
    __syncthreads();

    xmax = 0.0;
    imax = 0;
    sgnmax = 1.0;
    for (t = 0; t < 10; t++)
        if (abs(sWup[t]) > xmax) {
            xmax = abs(sWup[t]);
            imax = t;
            sgnmax = copysign(1.0, sWup[t]);
        }

    // Interpolate by imax
    tid = threadIdx.x;
    for (k = 0; k < Nrank; k++) {
        xshift = 0.0;
        for (t = 0; t < nt0; t++)
            xshift += B[tid + t * nt0 + nt0 * nt0 * imax] * sW[t + k * nt0];
        if (k == 0)
            xshift = -xshift * sgnmax;
        W[tid + bid * nt0 + k * nt0 * Nfilt] = xshift;
    }
}