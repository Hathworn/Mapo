#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 5000, NrankMax = 6;

__global__ void extract_snips2(const double *Params, const float *err, const int *st, const int *id, const int *counter, const int *kk, const int *iC, const float *W, float *WU){

    int nt0, tidx, tidy, bid, ind, icl, Nchan, Nmax, Nsum, NchanNear;

    nt0       = (int) Params[4];
    Nchan     = (int) Params[9];
    Nsum      = (int) Params[13];
    NchanNear = (int) Params[10];

    tidx = threadIdx.x;
    bid  = blockIdx.x;

    Nmax = min(maxFR, counter[1]);

    for(ind = 0; ind < Nmax; ind++) {
        if (id[ind] == bid) {
            tidy = threadIdx.y;
            icl = kk[st[ind]];
            int icl_mod_Nsum = icl % Nsum;  // Compute once and reuse
            float scale_factor = sqrt(err[st[ind]] / (1.0f + icl_mod_Nsum));  // Compute scale factor once
            while (tidy < (1 + icl_mod_Nsum)) {
                // Optimizing memory indices and calculations
                int offset = tidy + bid * NchanNear;
                WU[tidx + iC[offset] * nt0 + nt0 * Nchan * ind] = 
                    scale_factor * W[tidx + nt0 * (icl / Nsum)];
                tidy += blockDim.y;
            }
        }
    }
}