#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
__global__ void bestFilter(const double *Params, const bool *match, const int *iC, const int *call, const float *cmax, int *id, float *cx) {
    // Define shared memory to cache match and cmax values for better memory access
    extern __shared__ bool shared_match[];
    float max_running = 0.0f;

    int Nspikes = (int) Params[0];
    int Nfilters = (int) Params[2];
    int Nchan = (int) Params[7];
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int Nthreads = blockDim.x;
    int Nblocks = gridDim.x;

    int tind = tid + bid * Nthreads;
    
    // Calculate number of match entries each thread should handle
    int match_offset = tid * Nchan;
    if (match_offset < Nfilters * Nchan) {
        for (int k = 0; k < Nchan; k++) {
             shared_match[match_offset + k] = match[match_offset + k];
        }
    }
    __syncthreads(); // Make sure all threads have cached their data

    while (tind < Nspikes) {
        max_running = 0.0f;
        id[tind] = 0;
        int my_chan = call[tind];

        // Iterate over filters
        for (int ind = 0; ind < Nfilters; ind++) {
            if (shared_match[my_chan + ind * Nchan]) {
                float cmax_val = cmax[tind + ind * Nspikes];
                if (cmax_val > max_running) { // Update max_running if a new maximum is found
                    id[tind] = ind;
                    max_running = cmax_val;
                }
            }
        }

        cx[tind] = max_running;
        tind += Nblocks * Nthreads; // Move to the next spike index in CUDA grid
    }
}