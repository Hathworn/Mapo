#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024,  NrankMax = 3, nt0max = 71, NchanMax = 1024;

//////////////////////////////////////////////////////////////////////////////////////////

//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
//////////////////////////////////////////////////////////////////////////////////////////
__global__ void getwtw(const double *Params, const double *dWU, double *wtw) {
    int nt0, tidx, tidy, bid, Nchan, k;
    double x;

    nt0   = __double2int_rd(Params[4]);  // Convert to int using a more optimal method
    Nchan = __double2int_rd(Params[9]);

    tidx = threadIdx.x;
    tidy = threadIdx.y;

    bid = blockIdx.x;

    int offset = bid * Nchan * nt0;
    int stride = blockDim.y;

    while (tidy < nt0) {
        x = 0.0;
        for (k = 0; k < Nchan; k++) {
            int index = k * nt0 + offset;
            x += dWU[tidx + index] * dWU[tidy + index];
        }
        wtw[tidx + tidy * nt0 + bid * nt0 * nt0] = x;
        tidy += stride;  // Increment by blockDim.y for better utilization
    }
}