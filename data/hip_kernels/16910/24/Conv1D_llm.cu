#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
const int  Nthreads = 1024, maxFR = 10000, NrankMax = 3, nt0max=81, NchanMax = 17;

__global__ void Conv1D(const double *Params, const float *data, const float *W, float *conv_sig) {
    extern __shared__ float sW_sdata[];
    volatile float* sW = sW_sdata;
    volatile float* sdata = sW + 81 * NrankMax;

    float x, y;
    int tid, tid0, bid, i, nid, Nrank, NT, nt0;

    tid = threadIdx.x;
    bid = blockIdx.x;

    NT = (int) Params[0];
    nt0 = (int) Params[3];
    Nrank = (int) Params[6];

    // Preload weights into shared memory
    if (tid < nt0 * Nrank) {
        sW[tid] = W[tid];
    }
    __syncthreads();

    tid0 = 0;
    while (tid0 < NT - Nthreads - nt0 + 1) {
        // Load data into shared memory
        if (tid < nt0) {
            sdata[tid] = data[tid0 + tid + NT * bid];
        }
        sdata[tid + nt0] = data[tid0 + tid + nt0 + NT * bid];
        __syncthreads();

        // Initialize accumulator
        x = 0.0f;
        for (nid = 0; nid < Nrank; nid++) {
            y = 0.0f;
            #pragma unroll 4
            for (i = 0; i < nt0; i++) {
                y += sW[i + nid * nt0] * sdata[i + tid];
            }
            x += y * y;
        }

        // Write result to global memory
        conv_sig[tid0 + tid + NT * bid] = sqrtf(x);

        tid0 += Nthreads;
        __syncthreads();
    }
}