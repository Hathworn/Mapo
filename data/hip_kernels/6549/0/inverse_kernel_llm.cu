#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inverse_kernel(double* d_y, double* d_x) {
    // Use local variable to avoid global memory read latency 
    double x_local = d_x[0]; 
    // Prevent division by zero
    if (x_local != 0.0) {
        d_y[0] = 1.0 / x_local;
    } else {
        d_y[0] = 0.0; // Handle zero division safely
    }
}