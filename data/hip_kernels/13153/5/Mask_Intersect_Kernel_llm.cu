#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Mask_Intersect_Kernel(int* A, int* B, int* devOut, int numElements) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure the thread only processes valid index
    if (idx < numElements) {
        devOut[idx] = A[idx] * B[idx];
    }
}