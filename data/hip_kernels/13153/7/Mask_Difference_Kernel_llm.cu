#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Mask_Difference_Kernel(int* A, int* B, int* devOut) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Simplified without conditional branching
    devOut[idx] = (A[idx] != B[idx]);
}