#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Define a macro for handling warp-level reduction
#define WARP_SIZE 32

__global__ void Image_SumReduceStep_Kernel(int* devBufIn, int* devBufOut, int lastBlockSize)
{
    __shared__ int sharedMem[512];
    int globalIdx = 512 * blockIdx.x + threadIdx.x;
    int localIdx = threadIdx.x;

    // Initialize shared memory to zero
    sharedMem[localIdx] = (localIdx < 256 && blockIdx.x == gridDim.x - 1 && localIdx >= lastBlockSize) ? 0 : devBufIn[globalIdx];
    sharedMem[localIdx + 256] = (localIdx < 256 && blockIdx.x == gridDim.x - 1 && localIdx + 256 >= lastBlockSize) ? 0 : devBufIn[globalIdx + 256];

    __syncthreads();

    // Reduce using a single shared memory array, avoiding switching
    for (int stride = 256; stride > WARP_SIZE; stride >>= 1) {
        if (localIdx < stride / 2) {
            sharedMem[localIdx] += sharedMem[localIdx + stride / 2];
        }
        __syncthreads();
    }

    // Warp-level reduction
    if (localIdx < WARP_SIZE) {
        for (int stride = WARP_SIZE; stride > 0; stride >>= 1) {
            sharedMem[localIdx] += sharedMem[localIdx + stride];
            __syncthreads();
        }
    }

    // Write final result to global memory
    if (localIdx == 0) {
        devBufOut[blockIdx.x] = sharedMem[0];
    }
}