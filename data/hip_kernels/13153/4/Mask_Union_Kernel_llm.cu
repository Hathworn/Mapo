#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Mask_Union_Kernel(int* A, int* B, int* devOut) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use ternary operator for concise conditional assignment
    devOut[idx] = (A[idx] + B[idx] > 0) ? 1 : 0;
}