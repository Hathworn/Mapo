#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Mask_Subtract_Kernel(int* A, int* B, int* devOut)
{
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Use ternary operator for conditional assignment
    devOut[idx] = (B[idx] == 0) ? A[idx] : 0;
}