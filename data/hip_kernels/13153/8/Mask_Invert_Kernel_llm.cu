#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Mask_Invert_Kernel(int* A, int* devOut)
{
    // Calculate the global index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input value to register to reduce global memory accesses
    int input_value = A[idx];

    // Perform computation and store result
    devOut[idx] = 1 - input_value;
}