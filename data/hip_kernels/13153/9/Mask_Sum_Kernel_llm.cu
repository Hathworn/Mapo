#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Mask_Sum_Kernel(int* A, int valCount, int* scalarOut)
{
    const int localIdx = threadIdx.x;
    const int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        // Synchronize threads to ensure all additions are completed before proceeding
        __syncthreads();
        if (localIdx < stride)
        {
            A[globalIdx] += A[globalIdx + stride];
        }
    }

    // The first thread writes the result of this block to the output
    if (localIdx == 0)
    {
        A[blockIdx.x] = A[globalIdx];
    }
    
    // Synchronize before moving to the final reduction step
    __syncthreads();

    if (globalIdx == 0)
    {
        int sum = 0;
        for (int i = 0; i < gridDim.x; ++i)
        {
            sum += A[i * blockDim.x];
        }
        scalarOut[0] = sum;
    }
}