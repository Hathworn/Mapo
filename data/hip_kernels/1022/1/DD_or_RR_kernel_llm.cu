#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// 2-point angular correlation

const int BLOCKSIZE = 256;
const int ROWSPERTHREAD = 256;

// Columns are D and rows are R

// All computation in single-precision

__global__ void DD_or_RR_kernel(int nCols, int nRows, float *arr, unsigned long long int *gHist) {
    // Compute global thread x and row offset
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * ROWSPERTHREAD;
    
    // Check if within bounds
    if (x < nCols && y + ROWSPERTHREAD > x) {
        // Optimized use of shared memory for histogram
        __shared__ unsigned int sHist[720];

        // Initialize shared memory using thread block instead of single thread for better efficiency
        for (int i = threadIdx.x; i < 720; i += blockDim.x) {
            sHist[i] = 0;
        }

        __syncthreads();

        // Calculate local values based on offset
        float asc1 = arr[x * 2];
        float dec1 = arr[x * 2 + 1];
        int offset = max(x - y + 1, 0);
        int nElements = min(nRows - y, ROWSPERTHREAD);

        for (int j = offset; j < nElements; j++) {
            float asc2 = arr[(y + j) * 2];
            float dec2 = arr[(y + j) * 2 + 1];
            float tmp = sinf(dec1) * sinf(dec2) + cosf(dec1) * cosf(dec2) * cosf(asc1 - asc2);
            tmp = fminf(tmp, 1.0f);
            tmp = fmaxf(tmp, -1.0f);
            float radianResult = acosf(tmp);
            float degreeResult = radianResult * 180.0f / 3.14159f;
            int resultIndex = floor(degreeResult * 4.0f);

            // Increment using atomic operations for shared histogram
            atomicAdd(&sHist[resultIndex], 2);
        }

        __syncthreads();

        // Use all threads to write back to global memory for reduced contention
        for (int i = threadIdx.x; i < 720; i += blockDim.x) {
            atomicAdd(&gHist[i], sHist[i]);
        }
    }
}