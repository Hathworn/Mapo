#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// 2-point angular correlation

const int BLOCKSIZE = 256;
const int ROWSPERTHREAD = 256;

__global__ void DR_kernel(int nCols, int nRows, float *D, float *R, unsigned long long int *gHist) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * ROWSPERTHREAD;

    if (x < nCols) {
        __shared__ unsigned int sHist[720];

        // Initialize shared memory using all threads (faster parallel initialization)
        for (int i = threadIdx.x; i < 720; i += blockDim.x) {
            sHist[i] = 0;
        }

        __syncthreads();

        float asc1 = D[x * 2];
        float dec1 = D[x * 2 + 1];

        int nElements = min(nRows - y, ROWSPERTHREAD);

        for (int j = 0; j < nElements; j++) {
            float asc2 = R[(y + j) * 2];
            float dec2 = R[(y + j) * 2 + 1];

            float tmp = sinf(dec1) * sinf(dec2) + cosf(dec1) * cosf(dec2) * cosf(asc1 - asc2);
            tmp = fminf(fmaxf(tmp, -1.0f), 1.0f); // Combine clamping operations

            float radianResult = acosf(tmp);
            float degreeResult = radianResult * 180.0f / 3.14159f;

            int resultIndex = floor(degreeResult * 4.0f);

            atomicAdd(&sHist[resultIndex], 1);
        }

        __syncthreads();

        // Use all threads to update the global histogram (faster parallel reduction)
        for (int i = threadIdx.x; i < 720; i += blockDim.x) {
            atomicAdd(&gHist[i], sHist[i]);
        }
    }
}