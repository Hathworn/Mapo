#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define Tile_size 16  // Increase tile size for better occupancy

// Function To handle any errors occurred in the function calls
__global__ void matrixMultiplyShared(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns)
{
    __shared__ float sA[Tile_size][Tile_size];   // Tile size to store elements in shared memory
    __shared__ float sB[Tile_size][Tile_size];

    int Row = blockDim.y * blockIdx.y + threadIdx.y;  // Thread row index
    int Col = blockDim.x * blockIdx.x + threadIdx.x;  // Thread column index
    float Cvalue = 0.0;

    for (int k = 0; k < (((numAColumns - 1) / Tile_size) + 1); k++)
    {
        // Load data into shared memory if within bounds
        if ((Row < numARows) && (threadIdx.x + (k * Tile_size)) < numAColumns)
        {
            sA[threadIdx.y][threadIdx.x] = A[(Row * numAColumns) + threadIdx.x + (k * Tile_size)];
        }
        else
        {
            sA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if ((Col < numBColumns) && (threadIdx.y + k * Tile_size) < numBRows)
        {
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k * Tile_size) * numBColumns + Col];
        }
        else
        {
            sB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();  // Ensure all threads have loaded their data

        // Perform computation
        #pragma unroll
        for (int j = 0; j < Tile_size; ++j)
        {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }

        __syncthreads();  // Ensure all threads have completed computation before loading new data
    }

    // Store the result in global memory
    if (Row < numCRows && Col < numCColumns)
    {
        C[Row * numCColumns + Col] = Cvalue;
    }
}