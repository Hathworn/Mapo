#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" 
#include "includes.h"

// Optimize the kernel by using grid-stride loop to handle more data if needed
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (idx == 0) { // Only the first thread in the grid carries out the division
        *accuracy /= N;
    }
}