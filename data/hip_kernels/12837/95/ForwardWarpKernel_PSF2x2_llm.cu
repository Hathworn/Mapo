#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate coordinates for the corners
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    int tx = (int) px;
    int ty = (int) py;
    float value = src[image_row_offset + j];
    float weight;

    // Helper lambda to update dst and normalization_factor
    auto update_pixel = [&](int x, int y, float weight) {
        if (x >= 0 && x < w && y >= 0 && y < h) {
            int idx = y * image_stride + x;
            atomicAdd(dst + idx, value * weight);
            atomicAdd(normalization_factor + idx, weight);
        }
    };

    // Fill pixels using calculated weights
    update_pixel(tx, ty, dx * dy);
    update_pixel(tx - 1, ty, (1.0f - dx) * dy);
    update_pixel(tx - 1, ty - 1, (1.0f - dx) * (1.0f - dy));
    update_pixel(tx, ty - 1, dx * (1.0f - dy));
}