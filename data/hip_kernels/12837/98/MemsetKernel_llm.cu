#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global thread index
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary conditions
    if (i < h && j < w)
    {
        // Compute linear array index and set value
        int pos = i * w + j;
        image[pos] = value;
    }
}