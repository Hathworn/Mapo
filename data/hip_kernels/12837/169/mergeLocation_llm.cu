#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate point index for this thread
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the point index is within bounds
    if (ptidx < npoints)
    {
        // Load location data from global memory into a register
        short2 loc = __ldg(&loc_[ptidx]);

        // Perform the scaling and store results in global memory
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}