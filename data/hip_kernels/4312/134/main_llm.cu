#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "upsampleFilterForegroundMaskKernel_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    hipTextureObject_t subsampled_mask = 1;
unsigned upsample_rows = 1;
unsigned upsample_cols = 1;
unsigned sample_rate = 1;
const float sigma = 1;
hipSurfaceObject_t upsampled_mask = 1;
hipSurfaceObject_t filter_mask = 2;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
upsampleFilterForegroundMaskKernel<<<gridBlock, threadBlock>>>(subsampled_mask,upsample_rows,upsample_cols,sample_rate,sigma,upsampled_mask,filter_mask);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
upsampleFilterForegroundMaskKernel<<<gridBlock, threadBlock>>>(subsampled_mask,upsample_rows,upsample_cols,sample_rate,sigma,upsampled_mask,filter_mask);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
upsampleFilterForegroundMaskKernel<<<gridBlock, threadBlock>>>(subsampled_mask,upsample_rows,upsample_cols,sample_rate,sigma,upsampled_mask,filter_mask);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}