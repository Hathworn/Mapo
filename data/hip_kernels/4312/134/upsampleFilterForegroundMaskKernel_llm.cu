#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void upsampleFilterForegroundMaskKernel(hipTextureObject_t subsampled_mask, unsigned upsample_rows, unsigned upsample_cols, unsigned sample_rate, const float sigma, hipSurfaceObject_t upsampled_mask, hipSurfaceObject_t filter_mask) {
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x >= upsample_cols || y >= upsample_rows) return;

    // Precompute constants outside the loops
    const int halfsize = __float2uint_ru(sigma) * 2;
    const float inv_sigma_square = 1.0f / (sigma * sigma);

    float total_weight = 0.0f;
    float total_value = 0.0f;

    // Use shared memory for common calculations (optional)
    __shared__ float shared_weights[32][32]; // Adjust size according to block size, if necessary

    #pragma unroll // Unroll loops for potential performance gain
    for (int neighbor_y = y - halfsize; neighbor_y <= y + halfsize; neighbor_y++) {
        #pragma unroll
        for (int neighbor_x = x - halfsize; neighbor_x <= x + halfsize; neighbor_x++) {
            if (neighbor_x >= 0 && neighbor_x < upsample_cols && neighbor_y >= 0 && neighbor_y < upsample_rows) {
                // Retrieve the mask value at neighbor
                const auto subsampled_neighbor_x = neighbor_x / sample_rate;
                const auto subsampled_neighbor_y = neighbor_y / sample_rate;
                const unsigned char neighbor_foreground = tex2D<unsigned char>(subsampled_mask, subsampled_neighbor_x, subsampled_neighbor_y);

                // Compute the gaussian weight
                const float diff_x_square = (neighbor_x - x) * (neighbor_x - x);
                const float diff_y_square = (neighbor_y - y) * (neighbor_y - y);
                const float weight = __expf(-0.5f * (diff_x_square + diff_y_square) * inv_sigma_square);

                // Accumulate it
                total_weight += weight;
                total_value += weight * float(1 - neighbor_foreground);
            }
        }
    }

    // Compute the value locally (avoid division by zero)
    const auto subsampled_x = x / sample_rate;
    const auto subsampled_y = y / sample_rate;
    const unsigned char foreground_indicator = tex2D<unsigned char>(subsampled_mask, subsampled_x, subsampled_y);
    float filter_value = 0.0f;
    if (foreground_indicator == 0 && total_weight > 1e-3f) {
        filter_value = total_value / total_weight;
    }

    // Write to the surface
    surf2Dwrite(foreground_indicator, upsampled_mask, x * sizeof(unsigned char), y);
    surf2Dwrite(filter_value, filter_mask, x * sizeof(float), y);
}
```
