#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "clipFilterDepthKernel_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    hipTextureObject_t raw_depth = 1;
const unsigned clip_img_rows = 1;
const unsigned clip_img_cols = 1;
const unsigned clip_near = 1;
const unsigned clip_far = 1;
const float sigma_s_inv_square = 1;
const float sigma_r_inv_square = 1;
hipSurfaceObject_t filter_depth = 2;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
clipFilterDepthKernel<<<gridBlock, threadBlock>>>(raw_depth,clip_img_rows,clip_img_cols,clip_near,clip_far,sigma_s_inv_square,sigma_r_inv_square,filter_depth);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
clipFilterDepthKernel<<<gridBlock, threadBlock>>>(raw_depth,clip_img_rows,clip_img_cols,clip_near,clip_far,sigma_s_inv_square,sigma_r_inv_square,filter_depth);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
clipFilterDepthKernel<<<gridBlock, threadBlock>>>(raw_depth,clip_img_rows,clip_img_cols,clip_near,clip_far,sigma_s_inv_square,sigma_r_inv_square,filter_depth);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}