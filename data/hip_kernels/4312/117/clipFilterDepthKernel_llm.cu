#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clipFilterDepthKernel(hipTextureObject_t raw_depth, const unsigned clip_img_rows, const unsigned clip_img_cols, const unsigned clip_near, const unsigned clip_far, const float sigma_s_inv_square, const float sigma_r_inv_square, hipSurfaceObject_t filter_depth) {
    // Parallel over the clipped image
    const auto x = threadIdx.x + blockDim.x * blockIdx.x;
    const auto y = threadIdx.y + blockDim.y * blockIdx.y;

    if (y >= clip_img_rows || x >= clip_img_cols) return;

    // Compute the center on raw depth
    constexpr auto half_width = 5; // Use constexpr for constants
    const auto raw_x = x + boundary_clip;
    const auto raw_y = y + boundary_clip;
    const unsigned short center_depth = tex2D<unsigned short>(raw_depth, raw_x, raw_y);

    // Use shared memory for window depth values
    __shared__ unsigned short window_depth[(2 * half_width + 1) * (2 * half_width + 1)];
    // Load the window depth values into shared memory
    const auto local_x = threadIdx.x;
    const auto local_y = threadIdx.y;

    for (int dy = -half_width; dy <= half_width; ++dy) {
        for (int dx = -half_width; dx <= half_width; ++dx) {
            const auto offset_y = raw_y + dy;
            const auto offset_x = raw_x + dx;
            const auto local_idx = (dy + half_width) * (2 * half_width + 1) + (dx + half_width);
            window_depth[local_idx] = tex2D<unsigned short>(raw_depth, offset_x, offset_y);
        }
    }
    __syncthreads(); // Ensure all threads have loaded their values

    // Iterate over the window using loaded values
    float sum_all = 0.0f;
    float sum_weight = 0.0f;

    for (int dy = -half_width; dy <= half_width; ++dy) {
        for (int dx = -half_width; dx <= half_width; ++dx) {
            const auto local_idx = (dy + half_width) * (2 * half_width + 1) + (dx + half_width);
            const unsigned short depth = window_depth[local_idx];
            const float depth_diff2 = (depth - center_depth) * (depth - center_depth);
            const float pixel_diff2 = dx * dx + dy * dy;
            const float this_weight = (depth > 0) * expf(-sigma_s_inv_square * pixel_diff2) * expf(-sigma_r_inv_square * depth_diff2);
            sum_weight += this_weight;
            sum_all += this_weight * depth;
        }
    }

    // Put back to the filtered depth
    unsigned short filtered_depth_value = __float2uint_rn(sum_all / sum_weight);
    if (filtered_depth_value < clip_near || filtered_depth_value > clip_far) filtered_depth_value = 0;
    surf2Dwrite(filtered_depth_value, filter_depth, x * sizeof(unsigned short), y);
}