#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void filterDensityMapKernel(hipTextureObject_t density_map, unsigned rows, unsigned cols, hipSurfaceObject_t filter_density_map) {
    const auto x = threadIdx.x + blockIdx.x * blockDim.x;
    const auto y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x >= cols || y >= rows) return;

    const auto half_width = 5;
    const float center_density = tex2D<float>(density_map, x, y);

    // Initialize sums as registers for better performance
    float sum_all = 0.0f;
    float sum_weight = 0.0f;

    // Optimize window search with combined loop to reduce redundant calculations
    for (int dy = -half_width; dy <= half_width; dy++) {
        int y_idx = y + dy;
        for (int dx = -half_width; dx <= half_width; dx++) {
            int x_idx = x + dx;
            const float density = tex2D<float>(density_map, x_idx, y_idx);
            const float value_diff2 = (center_density - density) * (center_density - density);
            const float pixel_diff2 = dx * dx + dy * dy;
            const float this_weight = (density > 0.0f) * expf(-0.04f * pixel_diff2) * expf(-100.0f * value_diff2);
            sum_weight += this_weight;
            sum_all += this_weight * density;
        }
    }

    // Calculate filter value and clip it to a suitable range
    float filter_density_value = sum_all / sum_weight;
    filter_density_value = fminf(fmaxf(filter_density_value, 0.0f), 1.0f);

    surf2Dwrite(filter_density_value, filter_density_map, x * sizeof(float), y);
}