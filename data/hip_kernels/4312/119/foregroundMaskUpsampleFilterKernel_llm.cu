#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void foregroundMaskUpsampleFilterKernel(hipTextureObject_t subsampled_mask, unsigned upsample_rows, unsigned upsample_cols, const float sigma, hipSurfaceObject_t upsampled_mask, hipSurfaceObject_t filter_mask) {
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;

    // Return early if out of bounds
    if (x >= upsample_cols || y >= upsample_rows) return;

    const int halfsize = __float2uint_ru(sigma) * 2;
    float total_weight = 0.0f;
    float total_value = 0.0f;

    const auto subsampled_x = x / crf_subsample_rate;
    const auto subsampled_y = y / crf_subsample_rate;
    const unsigned char foreground_indicator = tex2D<unsigned char>(subsampled_mask, subsampled_x, subsampled_y);

    // Combine bounds check with accumulation
    for (int offset_y = -halfsize; offset_y <= halfsize; offset_y++) {
        const int neighbor_y = y + offset_y;
        for (int offset_x = -halfsize; offset_x <= halfsize; offset_x++) {
            const int neighbor_x = x + offset_x;

            // Check if neighbor is within bounds
            if (neighbor_x >= 0 && neighbor_x < upsample_cols && neighbor_y >= 0 && neighbor_y < upsample_rows) {
                const auto subsampled_neighbor_x = neighbor_x / crf_subsample_rate;
                const auto subsampled_neighbor_y = neighbor_y / crf_subsample_rate;

                // Retrieve the mask value at neighbor
                const unsigned char neighbor_foreground = tex2D<unsigned char>(subsampled_mask, subsampled_neighbor_x, subsampled_neighbor_y);

                // Compute the gaussian weight
                const float diff_x_square = offset_x * offset_x;
                const float diff_y_square = offset_y * offset_y;
                const float weight = __expf(0.5f * (diff_x_square + diff_y_square) / (sigma * sigma));
                
                // Accumulate the weight and value
                total_weight += weight;
                total_value += weight * float(1 - neighbor_foreground);
            }
        }
    }

    float filter_value = 0.0f;
    if (foreground_indicator == 0) {
        filter_value = total_value / (total_weight + 1e-3f);
    }

    // Write to the surfaces
    surf2Dwrite(foreground_indicator, upsampled_mask, x * sizeof(unsigned char), y);
    surf2Dwrite(filter_value, filter_mask, x * sizeof(float), y);
}