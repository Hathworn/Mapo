#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void markValidIndexMapPixelKernel(hipTextureObject_t index_map, int validity_halfsize, unsigned img_rows, unsigned img_cols, unsigned char* flatten_validity_indicator) {
    const auto x_center = threadIdx.x + blockDim.x * blockIdx.x;
    const auto y_center = threadIdx.y + blockDim.y * blockIdx.y;
    if (x_center >= img_cols || y_center >= img_rows) return;
    const auto offset = x_center + y_center * img_cols;

    // Only depend on this pixel
    if (validity_halfsize <= 0) {
        const auto surfel_index = tex2D<unsigned>(index_map, x_center, y_center);
        flatten_validity_indicator[offset] = (surfel_index != 0xFFFFFFFF);
        return;
    }

    // Perform a window search when halfsize is at least 1
    unsigned char validity = 1;
    for (auto y = max(0, y_center - validity_halfsize); y <= min((int)img_rows - 1, y_center + validity_halfsize); y++) {
        for (auto x = max(0, x_center - validity_halfsize); x <= min((int)img_cols - 1, x_center + validity_halfsize); x++) {
            if (tex2D<unsigned>(index_map, x, y) == 0xFFFFFFFF) {
                validity = 0;
                break; // Exit early if invalid pixel found
            }
        }
        if (validity == 0) break; // No need to continue checking
    }

    // Save it
    flatten_validity_indicator[offset] = validity;
}