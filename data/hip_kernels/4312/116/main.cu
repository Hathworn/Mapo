#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "markValidIndexMapPixelKernel.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    hipTextureObject_t index_map = 1;
int validity_halfsize = XSIZE*YSIZE;
unsigned img_rows = 1;
unsigned img_cols = 1;
unsigned char *flatten_validity_indicator = NULL;
hipMalloc(&flatten_validity_indicator, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
markValidIndexMapPixelKernel<<<gridBlock, threadBlock>>>(index_map,validity_halfsize,img_rows,img_cols,flatten_validity_indicator);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
markValidIndexMapPixelKernel<<<gridBlock, threadBlock>>>(index_map,validity_halfsize,img_rows,img_cols,flatten_validity_indicator);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
markValidIndexMapPixelKernel<<<gridBlock, threadBlock>>>(index_map,validity_halfsize,img_rows,img_cols,flatten_validity_indicator);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}