#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void markPotentialMatchedDepthPairKernel(hipTextureObject_t index_map, unsigned img_rows, unsigned img_cols, unsigned* reference_pixel_matched_indicator) {
    // Calculate global thread ID
    const auto x = blockIdx.x * blockDim.x + threadIdx.x;
    const auto y = blockIdx.y * blockDim.y + threadIdx.y;

    // Return if out of bounds
    if (x >= img_cols || y >= img_rows) return;

    // Compute offset for 1D array access
    const auto offset = y * img_cols + x;

    // Fetch the index value from the texture
    const auto surfel_index = tex2D<unsigned>(index_map, x, y);

    // Set indicator based on validity of surfel index
    reference_pixel_matched_indicator[offset] = (surfel_index != d_invalid_index) ? 1 : 0;
}