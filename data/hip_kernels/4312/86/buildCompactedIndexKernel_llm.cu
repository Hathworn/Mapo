#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void buildCompactedIndexKernel(const unsigned* valid_indicator, const unsigned table_size, unsigned* compacted_index) {
    const auto idx = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Ensure idx is within bounds before performing operations
    if (idx < table_size) {
        // Use shared memory to minimize global memory access
        __shared__ unsigned shared_valid_indicator[1024]; // Adjust size as needed
        shared_valid_indicator[threadIdx.x] = valid_indicator[idx];
        
        unsigned offset = 0xffffffffu;

        // Check the valid indicator using shared memory
        if (shared_valid_indicator[threadIdx.x] > 0) {
            offset = compacted_index[idx] - 1;
        }
        
        compacted_index[idx] = offset;
    }
}