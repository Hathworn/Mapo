#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel
__global__ void compactIndicatorToPixelKernel(const unsigned* candidate_pixel_indicator, const unsigned* prefixsum_indicator, unsigned img_cols, ushort2* compacted_pixels) {
    const auto idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Avoid unnecessary memory access by checking if the index is in bounds
    if (idx < img_cols * gridDim.x * blockDim.x && candidate_pixel_indicator[idx] > 0) {
        const auto offset = prefixsum_indicator[idx] - 1;
        const unsigned short x = idx % img_cols;
        const unsigned short y = idx / img_cols;
        compacted_pixels[offset] = make_ushort2(x, y);
    }
}