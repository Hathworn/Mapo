#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "compactIndicatorToPixelKernel.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    const unsigned *candidate_pixel_indicator = NULL;
hipMalloc(&candidate_pixel_indicator, XSIZE*YSIZE);
const unsigned *prefixsum_indicator = NULL;
hipMalloc(&prefixsum_indicator, XSIZE*YSIZE);
unsigned img_cols = 1;
ushort2 *compacted_pixels = NULL;
hipMalloc(&compacted_pixels, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
compactIndicatorToPixelKernel<<<gridBlock, threadBlock>>>(candidate_pixel_indicator,prefixsum_indicator,img_cols,compacted_pixels);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
compactIndicatorToPixelKernel<<<gridBlock, threadBlock>>>(candidate_pixel_indicator,prefixsum_indicator,img_cols,compacted_pixels);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
compactIndicatorToPixelKernel<<<gridBlock, threadBlock>>>(candidate_pixel_indicator,prefixsum_indicator,img_cols,compacted_pixels);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}