#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void assembleBinBlockCSRKernel(const unsigned matrix_size, const float* __restrict__ diagonal_blks, const float* __restrict__ nondiagonal_blks, const int* __restrict__ csr_rowptr, const unsigned* __restrict__ blkrow_offset, float* __restrict__ JtJ_data) {
    const unsigned row_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (row_idx >= matrix_size) return;

    int data_offset = csr_rowptr[row_idx];
    const unsigned blkrow_idx = row_idx / 6;
    const unsigned inblk_offset = row_idx % 6;

    // Use shared memory for diagonal blocks to reduce redundant global accesses
    __shared__ float shared_diagonal_blks[36];
    if (threadIdx.x < 36) {
        shared_diagonal_blks[threadIdx.x] = diagonal_blks[36 * blkrow_idx + threadIdx.x];
    }
    __syncthreads();

    // Fill the diagonal blocks with preloaded shared memory data
    for (unsigned k = 0; k < 6; k++, data_offset += bin_size) {
        JtJ_data[data_offset] = shared_diagonal_blks[inblk_offset + 6 * k];
    }

    // Fill the non-diagonal blocks without loop unrolling
    unsigned Iij_begin = blkrow_offset[blkrow_idx];
    const unsigned Iij_end = blkrow_offset[blkrow_idx + 1];
    for (; Iij_begin < Iij_end; Iij_begin++) {
        for (unsigned k = 0; k < 6; k++, data_offset += bin_size) {
            JtJ_data[data_offset] = nondiagonal_blks[36 * Iij_begin + inblk_offset + 6 * k];
        }
    }
}