#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void filterForegroundMaskKernel(hipTextureObject_t foreground_mask, unsigned mask_rows, unsigned mask_cols, const float sigma, hipSurfaceObject_t filter_mask) {
    // Calculate global thread coordinates
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;

    // Early exit if thread is out of bounds
    if (x >= mask_cols || y >= mask_rows) return;

    // Calculate parameters for window search
    const int halfsize = __float2uint_ru(sigma) * 2;
    float total_weight = 0.0f;
    float total_value = 0.0f;

    // Iterate over neighboring pixels within a window
    for (int neighbor_y = max(0, y - halfsize); neighbor_y <= min(mask_rows - 1, y + halfsize); neighbor_y++) {
        for (int neighbor_x = max(0, x - halfsize); neighbor_x <= min(mask_cols - 1, x + halfsize); neighbor_x++) {
            // Retrieve the mask value at neighbor
            const unsigned char neighbor_foreground = tex2D<unsigned char>(foreground_mask, neighbor_x, neighbor_y);

            // Compute the gaussian weight
            const float diff_x_square = (neighbor_x - x) * (neighbor_x - x);
            const float diff_y_square = (neighbor_y - y) * (neighbor_y - y);
            const float weight = __expf(0.5f * (diff_x_square + diff_y_square) / (sigma * sigma));

            // Accumulate weight and value
            total_weight += weight;
            total_value += weight * float(1 - neighbor_foreground);
        }
    }

    // Compute the value locally
    const unsigned char foreground_indicator = tex2D<unsigned char>(foreground_mask, x, y);
    float filter_value = 0.0f;
    if (foreground_indicator == 0) {
        filter_value = total_value / (total_weight + 1e-3f); // Avoid division by zero
    }

    // Write the computed filter value to the surface
    surf2Dwrite(filter_value, filter_mask, x * sizeof(float), y);
}