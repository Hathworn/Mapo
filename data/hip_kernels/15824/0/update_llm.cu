#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update(float *weights, float *grad, float lr, int N)
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if (x < N)
    {
        // Apply update and reset gradient in a single operation
        float gradient = grad[x];
        weights[x] -= lr * gradient;
        grad[x] = 0.0; // Reset to zero
    }
}