#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory to optimize transpose
__global__ void transposeNaive(float *odata, const float *idata, int idata_rows, int idata_cols)
{
    __shared__ float tile[TILE_SIZE][TILE_SIZE + 1]; // Avoid bank conflicts with padding

    int x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int y = blockIdx.y * TILE_SIZE + threadIdx.y;

    // Load data into shared memory
    if (y < idata_rows && x < idata_cols)
        tile[threadIdx.y][threadIdx.x] = idata[y * idata_cols + x];

    __syncthreads(); // Synchronize to ensure all data is loaded

    // Transpose within shared memory and store result
    x = blockIdx.y * TILE_SIZE + threadIdx.x; // Transpose block offset
    y = blockIdx.x * TILE_SIZE + threadIdx.y;

    if (y < idata_cols && x < idata_rows)
        odata[y * idata_rows + x] = tile[threadIdx.x][threadIdx.y];
}