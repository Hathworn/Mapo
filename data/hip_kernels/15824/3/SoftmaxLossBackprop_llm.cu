#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SoftmaxLossBackprop(const int *label, int num_labels, int batch_size, float *diff)
{
    // Calculate unique thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if the thread index exceeds the batch size
    if (idx >= batch_size) return;
    
    // Use label value directly in computation
    diff[idx * num_labels + label[idx]] -= 1.0f;
}