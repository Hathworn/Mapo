#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrixMultiplyNaive(float * A, float * B, float * C, int N, int K, int M)
{
    int Row = blockDim.y * blockIdx.y + threadIdx.y; // Calculate Row index for the thread
    int Col = blockDim.x * blockIdx.x + threadIdx.x; // Calculate Col index for the thread

    // Check if the thread is within matrix bounds
    if (Row < N && Col < M)
    {
        float Cvalue = 0.0f;
        // Optimize loop by using local variable as temporary storage
        for (int k = 0; k < K; ++k)
        {
            Cvalue += A[Row * K + k] * B[k * M + Col];
        }
        C[Row * M + Col] = Cvalue; // Store result in global memory
    }
}