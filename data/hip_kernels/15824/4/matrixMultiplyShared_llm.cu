#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_SIZE 32

__global__ void matrixMultiplyShared(float * A, float * B, float * C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns)
{
    __shared__ float sA[TILE_SIZE][TILE_SIZE]; // Shared memory tiles
    __shared__ float sB[TILE_SIZE][TILE_SIZE];

    int Row = blockDim.y * blockIdx.y + threadIdx.y; // Row index of C in global memory
    int Col = blockDim.x * blockIdx.x + threadIdx.x; // Column index of C in global memory
    float Cvalue = 0.0;

    for (int k = 0; k < (numAColumns + TILE_SIZE - 1) / TILE_SIZE; ++k)
    {
        // Collaborative loading of Tiles into shared memory
        if (Row < numARows && threadIdx.x + k * TILE_SIZE < numAColumns)
        {
            sA[threadIdx.y][threadIdx.x] = A[Row * numAColumns + threadIdx.x + k * TILE_SIZE];
        }
        else
        {
            sA[threadIdx.y][threadIdx.x] = 0.0;
        }

        if (Col < numBColumns && threadIdx.y + k * TILE_SIZE < numBRows)
        {
            sB[threadIdx.y][threadIdx.x] = B[(threadIdx.y + k * TILE_SIZE) * numBColumns + Col];
        }
        else
        {
            sB[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads(); // Ensure tiles are loaded

        // Perform the computation
        for (int j = 0; j < TILE_SIZE; ++j)
        {
            Cvalue += sA[threadIdx.y][j] * sB[j][threadIdx.x];
        }

        __syncthreads(); // Ensure computation uses same shared memory
    }

    // Write result to C
    if (Row < numCRows && Col < numCColumns)
    {
        C[Row * numCColumns + Col] = Cvalue;
    }
}