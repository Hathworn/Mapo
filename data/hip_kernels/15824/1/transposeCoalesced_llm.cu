#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeCoalesced(float *odata, const float *idata, int idata_rows, int idata_cols)
{
    __shared__ float tile[TILE_SIZE][TILE_SIZE+1];

    // Calculate indices
    int x = blockIdx.x * TILE_SIZE + threadIdx.x;
    int y = blockIdx.y * TILE_SIZE + threadIdx.y;

    // Load data into shared memory with coalescing
    if (x < idata_cols) {
        for (int j = 0; j < TILE_SIZE; j += BLOCK_SIZE) {
            if ((y + j) < idata_rows) {
                tile[threadIdx.y + j][threadIdx.x] = idata[(y + j) * idata_cols + x];
            }
        }
    }

    __syncthreads();

    // Swap block offset for transposing
    x = blockIdx.y * TILE_SIZE + threadIdx.x;
    y = blockIdx.x * TILE_SIZE + threadIdx.y;

    // Store transposed data from shared memory to global memory
    if (x < idata_rows) {
        for (int j = 0; j < TILE_SIZE; j += BLOCK_SIZE) {
            if ((y + j) < idata_cols) {
                odata[(y + j) * idata_rows + x] = tile[threadIdx.x][threadIdx.y + j];
            }
        }
    }
}