#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void opt_32to8Kernel(uint32_t *input, uint8_t* output, size_t length) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    // Check within bounds to avoid out-of-bounds memory access
    if (idx < length) {
        output[idx] = (input[idx] < UINT8_MAX) ? static_cast<uint8_t>(input[idx]) : UINT8_MAX;
    }

    // Remove unnecessary __syncthreads() for this simple assignment operation
}