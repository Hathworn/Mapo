#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_256_one_1024(float *A, float *B, float *bnBias, float *bnScale, float *C) {
    int tile = blockIdx.x, part = blockIdx.y, in_channel = threadIdx.x, line = threadIdx.y;
    int ind = line * 256 + in_channel;

    extern __shared__ float shared_[];
    float *weights = shared_ + 256 * 4, *output = weights + 256 * 32, *input = shared_;
    float *bias = output + 4 * 256, *scale = bias + 256;

    // Load data into shared memory
    input[ind] = A[tile * 1024 + ind];
    bias[in_channel] = bnBias[part * 256 + in_channel];
    scale[in_channel] = bnScale[part * 256 + in_channel];
    output[ind] = 0.0f;
    __syncthreads();

    // Loop optimization to improve memory coalescing and access patterns
    for (int k = 0; k < 256; k += 32) {
        for (int i = 0; i < 8; i++) {
            weights[ind + 1024 * i] = B[(k + i * 4 + line) * 1024 + part * 256 + in_channel];
        }
        __syncthreads();

        // Access shared memory efficiently
        float *A_start = input + k;
        float temp_sum = 0.0f;
        for (int p = 0; p < 32; p++) {
            temp_sum += A_start[line * 256 + p] * weights[in_channel + p * 256];
        }
        output[ind] += temp_sum;
        __syncthreads();
    }

    // Write final result to global memory
    float *C_start = C + tile * 4096 + part * 256;
    C_start[line * 1024 + in_channel] = scale[in_channel] * output[ind] + bias[in_channel];
}