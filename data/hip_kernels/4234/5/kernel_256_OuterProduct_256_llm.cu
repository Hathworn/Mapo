#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_256_OuterProduct_256(float *A, float *B, float *C) {
    int Tile = blockIdx.x, Part = blockIdx.y;
    int tX = threadIdx.x, tY = threadIdx.y;
    int c_input = tY * 256 + tX;
    int T_offset = (Tile << 12) + (Part << 11) + c_input;
    int B_offset = (Tile << 16) + c_input;

    extern __shared__ float shared_mem[];
    float *input = shared_mem;
    float *kernel = input + 2048;
    float *out = kernel + 8192;

    const int B_stride[32] = {
        0, 256, 512, 768, 1024, 1280, 1536, 1792, 
        2048, 2304, 2560, 2816, 3072, 3328, 3584, 3840, 
        4096, 4352, 4608, 4864, 5120, 5376, 5632, 5888, 
        6144, 6400, 6656, 6912, 7168, 7424, 7680, 7936
    };

    // Zero out partial results
    out[c_input] = 0.0f;
    out[c_input + 1024] = 0.0f;

    // Load inputs from global memory
    input[c_input] = A[T_offset];
    input[c_input + 1024] = A[T_offset + 1024];

    #pragma unroll
    for (int k = 0; k < 8; k++) {
        int B_start = B_offset + (k << 13);

        // Load kernel values from global memory
        #pragma unroll
        for (int i = 0; i < 8; ++i) {
            kernel[c_input + i * 1024] = B[B_start + i * 1024];
        }

        __syncthreads();

        float sum = 0, sum1 = 0;
        int y_tmp = (tY << 8) + (k << 5);
        int y_tmp1 = y_tmp + 1024;

        // Compute outer product
        #pragma unroll
        for (int j = 0; j < 32; j++) {
            float k_val = kernel[tX + B_stride[j]];
            sum += input[y_tmp + j] * k_val;
            sum1 += input[y_tmp1 + j] * k_val;
        }
        
        out[c_input] += sum;
        out[c_input + 1024] += sum1;

        __syncthreads();
    }

    // Store results to global memory
    C[T_offset] = out[c_input];
    C[T_offset + 1024] = out[c_input + 1024];
}