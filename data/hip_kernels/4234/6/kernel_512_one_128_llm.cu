#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_512_one_128(float *A, float *B, float *bnBias, float *bnScale, float *C) {
    int tile = blockIdx.x, in_channel = threadIdx.x, line = threadIdx.y;
    int ind = line * 128 + in_channel;

    extern __shared__ float shared_[];

    // Use shared memory efficiently
    float *weights = shared_ + 512 * 4, *output = weights + 128 * 64, *input = shared_;
    float *bias = output + 4 * 128, *scale = bias + 128;

    for (int i = 0; i < 4; i++)
        input[ind + i * 512] = A[tile * 2048 + i * 512 + ind];
    bias[in_channel] = bnBias[in_channel];
    scale[in_channel] = bnScale[in_channel];
    output[ind] = 0.0f;
    __syncthreads();

    // Loop unrolling for better performance
    for (int k = 0; k < 512; k += 64) {
        float *B_start = B + k * 128;
        for (int i = 0; i < 16; i++)
            weights[ind + i * 512] = B_start[i * 512 + ind];
        __syncthreads();

        float *A_start = input + k;
        float acc = 0.0f;
        #pragma unroll 8  // Unroll the loop for increased performance
        for (int p = 0; p < 64; p++) {
            acc += A_start[line * 512 + p] * weights[in_channel + p * 128];
        }
        output[ind] += acc;
        __syncthreads();
    }

    float *C_start = C + tile * 512;
    float res = scale[in_channel] * output[ind] + bias[in_channel];

    // Use max function for better readability and possibly performance improvements
    C_start[ind] = fmaxf(res, 0.0f);
}