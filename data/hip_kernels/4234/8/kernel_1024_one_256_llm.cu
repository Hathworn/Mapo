#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_1024_one_256(float *A, float *B, float *bnBias, float *bnScale, float *C) {
    // Calculate indices
    int tile = blockIdx.x, in_channel = threadIdx.x, line = threadIdx.y;
    int ind = line*256 + in_channel;

    // Shared memory declarations
    extern __shared__ float shared_[];
    float *weights = shared_ + 1024*4, *output = weights + 256*16, *input = shared_;
    float *bias = output + 4*256, *scale = bias + 256;

    // Efficiently loading A, biases & scales into shared memory
    for (int i = 0; i < 4; i++) {
        input[ind + i*1024] = A[tile*4096 + i*1024 + ind];
    }
    if (line == 0) {
        bias[in_channel] = bnBias[in_channel];
        scale[in_channel] = bnScale[in_channel];
    }
    
    output[ind] = 0.0f;
    __syncthreads();

    // Compute using shared weights and inputs
    for (int k = 0; k < 1024; k += 16) {
        float *B_start = B + k*256;
        for (int i = 0; i < 4; i++) {
            weights[ind + i*1024] = B_start[i*1024 + ind];
        }
        __syncthreads();

        float *A_start = input + k;
        for (int p = 0; p < 16; p++) {
            // Multiply-accumulate operation (using shared memory efficiently)
            output[ind] += A_start[line*1024 + p] * weights[in_channel + p*256];
        }
        __syncthreads();
    }

    // Write back results with ReLU activation
    if (line == 0) {
        float *C_start = C + tile*1024;
        float res = scale[in_channel] * output[ind] + bias[in_channel];
        C_start[ind] = res > 0 ? res : 0;
    }
}