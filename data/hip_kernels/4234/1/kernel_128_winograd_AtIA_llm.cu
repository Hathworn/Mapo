#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_128_winograd_AtIA(float *pInputs, float *pBiases, float *pScales, float *pOutputs) {
    int Tilex = blockIdx.x, Tiley = blockIdx.y, Iny = threadIdx.y, kz = blockIdx.z, Inx = threadIdx.x;
    int c_input = Inx * 6 + Iny;

    __shared__ float bias, scale;
    extern __shared__ float input[];

    // Preload bias and scale once per block
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        bias = pBiases[kz];
        scale = pScales[kz];
    }
    input[c_input] = pInputs[c_input * 16 * 128 + (Tilex * 4 + Tiley) * 128 + kz];
    __syncthreads();

    float tmp = 0;
    // Optimized calculation by unrolling loops and minimizing conditional branches
    switch(Inx) {
    case 0:
        tmp = input[Iny] + input[6 + Iny] + input[12 + Iny] + input[18 + Iny] + input[24 + Iny];
        break;
    case 1:
        tmp = input[6 + Iny] - input[12 + Iny] + 2 * input[18 + Iny] - 2 * input[24 + Iny];
        break;
    case 2:
        tmp = input[6 + Iny] + input[12 + Iny] + 4 * input[18 + Iny] + 4 * input[24 + Iny];
        break;
    case 3:
        tmp = input[6 + Iny] - input[12 + Iny] + 8 * input[18 + Iny] - 8 * input[24 + Iny] + input[30 + Iny];
        break;
    }
    __syncthreads();

    input[c_input] = tmp;
    __syncthreads();

    if (Inx > 3 || (Tilex == 3 && Inx > 1)) return;

    int x = Inx * 6;
    float o;
    switch(Iny) {
    case 0:
        o = scale * (input[x] + input[x + 1] + input[x + 2] + input[x + 3] + input[x + 4]) + bias;
        pOutputs[(((Tilex << 2) + 1 + Inx) * 16 + (Tiley << 2) + 1) * 128 + kz] = fmaxf(o, 0); // Use fmaxf for better performance
        break;
    case 1:
        o = scale * (input[x + 1] - input[x + 2] + 2 * input[x + 3] - 2 * input[x + 4]) + bias;
        pOutputs[(((Tilex << 2) + 1 + Inx) * 16 + (Tiley << 2) + 2) * 128 + kz] = fmaxf(o, 0);
        break;
    case 2:
        if (Tiley != 3) {
            o = scale * (input[x + 1] + input[x + 2] + 4 * input[x + 3] + 4 * input[x + 4]) + bias;
            pOutputs[(((Tilex << 2) + 1 + Inx) * 16 + (Tiley << 2) + 3) * 128 + kz] = fmaxf(o, 0);
        }
        break;
    case 3:
        if (Tiley != 3) {
            o = scale * (input[x + 1] - input[x + 2] + 8 * input[x + 3] - 8 * input[x + 4] + input[x + 5]) + bias;
            pOutputs[(((Tilex << 2) + 1 + Inx) * 16 + (Tiley << 2) + 4) * 128 + kz] = fmaxf(o, 0);
        }
        break;
    }
}