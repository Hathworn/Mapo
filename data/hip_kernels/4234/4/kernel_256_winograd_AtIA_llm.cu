#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_256_winograd_AtIA(float *pInputs, float *pBiases, float *pScales, float *pOutputs) {
    int Tilex = blockIdx.x, Tiley = blockIdx.y, Iny = threadIdx.y, kz = blockIdx.z, Inx = threadIdx.x;
    int c_input = Inx*6 + Iny;

    __shared__ float bias, scale;
    extern __shared__ float input[];

    // Load input and constants to shared memory
    input[c_input] = pInputs[c_input*16*256 + (Tilex*4+Tiley)*256 + kz];
    bias = pBiases[kz];
    scale = pScales[kz];
    __syncthreads();

    float tmp = 0;
    // Optimize computation with common subexpression elimination
    float input_6 = input[6+Iny];
    float input_12 = input[12+Iny];
    float input_18 = input[18+Iny];
    float input_24 = input[24+Iny];
    
    switch(Inx) {
    case 0:
        tmp = input[Iny] + input_6 + input_12 + input_18 + input_24;
        break;
    case 1:
        tmp = input_6 - input_12 + 2 * (input_18 - input_24);
        break;
    case 2:
        tmp = input_6 + input_12 + 4 * (input_18 + input_24);
        break;
    case 3:
        tmp = input_6 - input_12 + 8 * (input_18 - input_24) + input[30+Iny];
        break;
    }
    __syncthreads();

    input[c_input] = tmp;
    __syncthreads();

    // Simplify condition checks and avoid redundant computations
    if (Inx > 3 || (Tilex == 3 && Inx > 1)) return;

    int x = Inx * 6;
    float o;
    x += Iny == 0 ? 0 : 1; 

    const float input_x1 = input[x+1];
    const float input_x2 = input[x+2];
    const float input_x3 = input[x+3];
    const float input_x4 = input[x+4];

    switch(Iny) {
    case 0:
        o = scale * (input_x1 + input_x2 + input_x3 + input_x4 + input[x]) + bias;
        pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+1)*256 + kz] = max(o, 0.0f);
        break;
    case 1:
        o = scale * (input_x1 - input_x2 + 2 * (input_x3 - input_x4)) + bias;
        pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+2)*256 + kz] = max(o, 0.0f);
        break;
    case 2:
        if (Tiley != 3) {
            o = scale * (input_x1 + input_x2 + 4 * (input_x3 + input_x4)) + bias;
            pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+3)*256 + kz] = max(o, 0.0f);
        }
        break;
    case 3:
        if (Tiley != 3) {
            o = scale * (input_x1 - input_x2 + 8 * (input_x3 - input_x4) + input[x+5]) + bias;
            pOutputs[(((Tilex<<2)+1+Inx)*16 + (Tiley<<2)+4)*256 + kz] = max(o, 0.0f);
        }
        break;
    }
}