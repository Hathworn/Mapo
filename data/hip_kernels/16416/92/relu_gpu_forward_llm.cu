#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;

    // Process multiple elements per thread to improve memory throughput
    for (int i = tid; i < N; i += numThreads) {
        out[i] = in[i] > 0 ? in[i] : 0;
    }
}