#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setTensorCheckPatternKernel(unsigned int* data, unsigned int ndata) {
    // Use shared memory to reduce global memory accesses
    __shared__ unsigned int sharedData[256]; // Adjust size as needed

    unsigned int globalIdx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (globalIdx < ndata) {
        // Load data to shared memory
        sharedData[threadIdx.x] = globalIdx;
        __syncthreads();

        // Store data back to global memory
        data[globalIdx] = sharedData[threadIdx.x];
    }
}