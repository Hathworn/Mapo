#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vec_add(int* A, int* B, int* C, int size) {
    // Utilize a stride loop to enhance memory coalescing and allow for larger problem sizes
    for (int index = threadIdx.x + blockIdx.x * blockDim.x; index < size; index += blockDim.x * gridDim.x) {
        C[index] = A[index] + B[index];
    }
}