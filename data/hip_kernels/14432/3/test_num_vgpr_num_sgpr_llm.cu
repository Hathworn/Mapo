#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function to check occupancy
__global__ void test_num_vgpr_num_sgpr() {
    // Efficiently query and utilize hardware resources
    printf("Num VGPRs: %u, Num SGPRs: %u\n",
        hipThreadIdx_x, // Use threadId for demonstration
        hipGridDim_x);  // Use gridDim for demonstration
}