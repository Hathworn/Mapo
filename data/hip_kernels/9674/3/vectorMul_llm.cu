#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorMul(const float *A, const float *B, float *C, int numElements)
{
    // Use more descriptive variable names
    int globalIdx = blockDim.x * blockIdx.x + threadIdx.x;

    // Reduce the number of operations by storing values in the register
    if (globalIdx < numElements)
    {
        float a = A[globalIdx];
        float b = B[globalIdx];
        C[globalIdx] = a * b;
    }
}