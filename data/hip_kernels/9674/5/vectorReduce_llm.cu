#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorReduce(const float *global_input_data, float *global_output_data, const int numElements)
{
    __shared__ float sdata[10];
    __shared__ int sindice[10];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load elements into shared memory
    if (i < numElements) {
        sdata[tid] = global_input_data[i];
        sindice[tid] = tid;
    } else {
        sdata[tid] = FLT_MAX;  // Set maximum value for unused elements
        sindice[tid] = -1;     // Set invalid index for unused elements
    }
    __syncthreads();

    // Reduction process
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s && (i + s) < numElements) { // Ensure within range
            if (sdata[tid] > sdata[tid + s]) {
                sdata[tid] = sdata[tid + s];
                sindice[tid] = sindice[tid + s];
            }
        }
        __syncthreads();
    }

    // Write results to global memory
    if (tid == 0) {
        global_output_data[blockIdx.x * 2] = sdata[0];
        global_output_data[blockIdx.x * 2 + 1] = sindice[0];
    }
}