#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimizing global kernel function by introducing shared memory
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    __shared__ float shared_A[256]; // Assuming a maximum of 256 threads per block
    __shared__ float shared_B[256]; // Assuming a maximum of 256 threads per block

    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        // Load data into shared memory
        shared_A[threadIdx.x] = A[i];
        shared_B[threadIdx.x] = B[i];
        __syncthreads(); // Ensure all threads have loaded their data

        // Perform computation using shared memory
        C[i] = shared_A[threadIdx.x] + shared_B[threadIdx.x];
    }
}