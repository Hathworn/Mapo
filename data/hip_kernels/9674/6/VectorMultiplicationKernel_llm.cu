#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void VectorMultiplicationKernel(int *array, int arrayCount)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensures threads are not wasted by processing only valid indices
    for (int i = idx; i < arrayCount; i += blockDim.x * gridDim.x) {
        array[i] *= array[i];
    }
}