#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*
:copyright:
William B. Frank and Eric Beauce
:license:
GNU General Public License, Version 3
(https://www.gnu.org/licenses/gpl-3.0.en.html)
*/

// Optimized kernel for network correlation
__global__ void network_corr(float *templates, float *sum_square_template, int *moveout, float *data, float *weights, size_t step, size_t n_samples_template, size_t n_samples_data, size_t n_stations, size_t n_components, int chunk_offset, int chunk_size, float *cc_mat) {
    int idx = blockIdx.x / n_stations * blockDim.x + chunk_offset;
    int first_sample_block = idx * step;
    int s = blockIdx.x % n_stations;
    extern __shared__ float shared[];
    float *ss_template = &shared[0];
    float *templates_s = &shared[1];  // Adjust for single offset
    float *data_s = &shared[(n_samples_template / WARPSIZE + 1) * WARPSIZE + 1];

    for (int c = 0; c < n_components; c++) {
        if (weights[s * n_components + c] != 0.0f) {
            int cc_mat_offset = (first_sample_block / step + threadIdx.x - chunk_offset) * n_stations * n_components + s * n_components + c;
            int templates_offset = s * n_samples_template * n_components + c * n_samples_template;
            int sum_square_template_offset = s * n_components + c;
            int first_sample_trace = first_sample_block + moveout[s * n_components + c];
            int last_sample_trace = first_sample_trace + n_samples_template + threadIdx.x * step;
            int data_offset = s * n_samples_data * n_components + c * n_samples_data + first_sample_trace;
            float sum_square_data = 0.0f;
            float numerator = 0.0f;

            // Load template and data into shared memory
            int t_idx = threadIdx.x;
            if (t_idx == 0) {
                ss_template[0] = sum_square_template[sum_square_template_offset];
            }
            while (t_idx < n_samples_template) {
                templates_s[t_idx] = templates[templates_offset + t_idx];
                if ((first_sample_trace + t_idx) < n_samples_data) data_s[t_idx] = data[data_offset + t_idx];
                t_idx += blockDim.x;
            }
            while (t_idx < (blockDim.x * step + n_samples_template)) {
                if ((first_sample_trace + t_idx) < n_samples_data) data_s[t_idx] = data[data_offset + t_idx];
                t_idx += blockDim.x;
            }

            __syncthreads();  // Ensure shared memory is updated

            // Calculate correlation coefficient
            if (last_sample_trace < n_samples_data) {
                for (int i = 0; i < n_samples_template; i++) {
                    float data_sample = data_s[i + threadIdx.x * step];
                    numerator += data_sample * templates_s[i];
                    sum_square_data += data_sample * data_sample;
                }

                float denominator = sum_square_data * ss_template[0];
                if (cc_mat_offset < (chunk_size * n_stations * n_components)) {
                    // Prevent writing out of bounds
                    if (denominator > STABILITY_THRESHOLD) cc_mat[cc_mat_offset] = numerator * rsqrtf(denominator);
                }
            }
        }
    }
}