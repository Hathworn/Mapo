#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void double_value(double *x, double *y)
{
    // Use blockIdx.x and blockDim.x for wider parallel execution
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    y[idx] = 2.0 * x[idx]; // Avoid recomputing index; use '2.0' for clarity
}
```
