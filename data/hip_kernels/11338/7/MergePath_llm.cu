#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int satisfies(int i, int j, int *A, int *B)
{
    return (A[i] <= B[j]);
}

__global__ void MergePath(int *A, int *B, int* C, int *x, int *y, int n)
{
    int num_of_threads = blockDim.x;
    int idx = threadIdx.x;
    bool flag = false;

    // Initialize base case for the 0th index thread
    if (idx == 0)
    {
        x[idx] = 0;
        y[idx] = 0;
        flag = true;
    }

    int A_start = idx * (2 * n) / num_of_threads; // Calculate starting indices
    int B_start = max(0, A_start - (n - 1));
    A_start = min(n - 1, A_start);
    int length_of_array = (B_start == 0) ? (A_start + 1) : (n - B_start);

    int left = 0, right = length_of_array - 1;

    // Perform binary search to find the intersection point
    while (left <= right && !flag)
    {
        int mid = left + (right - left) / 2;
        int I = A_start - mid;
        int J = B_start + mid;

        if (!satisfies(I, J, A, B))
        {
            left = mid + 1;
        }
        else
        {
            if (J == 0 || I == n - 1 || !satisfies(I + 1, J - 1, A, B))
            {
                x[idx] = (I + 1);
                y[idx] = (J);
                flag = true;
            }
            else
            {
                right = mid;
            }
        }
    }

    if (!flag)
    {
        left--;
        x[idx] = (A_start - left);
        y[idx] = n;
    }
    __syncthreads();

    // Determine segment for merging
    int end_x = (idx == num_of_threads - 1) ? n : x[idx + 1];
    int end_y = (idx == num_of_threads - 1) ? n : y[idx + 1];
    int cur_x = x[idx];
    int cur_y = y[idx];
    int put_at = cur_x + cur_y;

    // Merge the sorted segments
    while (cur_x < end_x && cur_y < end_y)
    {
        if (A[cur_x] <= B[cur_y])
        {
            C[put_at++] = A[cur_x++];
        }
        else
        {
            C[put_at++] = B[cur_y++];
        }
    }

    // Copy remaining elements
    while (cur_x < end_x)
    {
        C[put_at++] = A[cur_x++];
    }
    while (cur_y < end_y)
    {
        C[put_at++] = B[cur_y++];
    }
}