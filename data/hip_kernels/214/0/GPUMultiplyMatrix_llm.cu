#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int NUMTHREADS = 1024;
int startNodeNumber;
int endNodeNumber;

typedef struct lList {
    int path[50];
    struct lList *next;
} lList;

__global__ void GPUMultiplyMatrix(long *matrix1, long *matrix2, int paths, int count) {
    int element = blockIdx.x * blockDim.x + threadIdx.x;
    if (element >= count * count) return; // Boundary check

    int col = element % count;
    int row = element / count;

    while (paths > 0) {
        long sum = 0;

        // Optimize using local variables
        for (int i = 0; i < count; i++) {
            long m1_val = matrix1[count * i + col];
            long m2_val = matrix2[row * count + i];
            sum += m1_val * m2_val;
        }

        // Store result directly to memory to avoid overwriting if multiple paths
        __syncthreads(); // Sync before writing
        matrix2[element] = sum;

        paths--;
    }
}