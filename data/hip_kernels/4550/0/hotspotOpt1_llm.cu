#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hotspotOpt1(float *p, float* tIn, float *tOut, float sdc, int nx, int ny, int nz, float ce, float cw, float cn, float cs, float ct, float cb, float cc)
{
    float amb_temp = 80.0;

    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    // Pre-compute repeated multiplications to reduce computation inside the loop
    int xy = nx * ny;
    int ij = i + j * nx;

    // Check if thread is within the valid range to avoid out-of-bound access
    if (i >= nx || j >= ny) return;

    // Performance measurement loop
    for(int run = 0; run < 100; run++){
        int c = ij;
        int W = (i == 0) ? c : c - 1;
        int E = (i == nx - 1) ? c : c + 1;
        int N = (j == 0) ? c : c - nx;
        int S = (j == ny - 1) ? c : c + nx;

        // Load values into registers to minimize global memory accesses
        float temp1, temp2, temp3;
        temp1 = temp2 = tIn[c];
        temp3 = tIn[c + xy];
        tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
                + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
        
        c += xy;
        W += xy;
        E += xy;
        N += xy;
        S += xy;

        // Main loop for layer processing
        #pragma unroll
        for (int k = 1; k < nz - 1; ++k) {
            temp1 = temp2;
            temp2 = temp3;
            temp3 = tIn[c + xy];
            tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
                    + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
            
            c += xy;
            W += xy;
            E += xy;
            N += xy;
            S += xy;
        }

        // Handle the last k iteration
        temp1 = temp2;
        temp2 = temp3;
        tOut[c] = cc * temp2 + cw * tIn[W] + ce * tIn[E] + cs * tIn[S]
                + cn * tIn[N] + cb * temp1 + ct * temp3 + sdc * p[c] + ct * amb_temp;
    }
    return;
}