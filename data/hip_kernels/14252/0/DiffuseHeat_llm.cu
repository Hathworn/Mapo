#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
#define CUDA_INPUT "input.txt"
#define CUDA_OUTPUT "cuda_output.txt"

int NUMPOINTS;
double ENDTIME;
double DT;
double ENDVALUES;

void InitialiseToZero(float* array);
__device__ void PrintPointsGPU(float* array, int size, double currentTime);
void PrintPointsCPU(float* array, double currentTime);
void ProcessOutput(float* array, int testCase, float time);
void CheckPoints(float* firstArray, float* secondArray);

__global__ void DiffuseHeat(float* currentPoints, float* nextPoints, const size_t size, double dx, double dt, const size_t endTime)
{
    unsigned int threadIndex = blockIdx.x * blockDim.x + threadIdx.x + 1;  // Calculate single expression for index
    if (threadIndex >= size - 1) return;  // Return if out of bounds to avoid unnecessary code execution

    double currentTime = 0.0;
    while (currentTime < endTime)
    {
        nextPoints[threadIndex] = currentPoints[threadIndex] + 0.25 * (
            currentPoints[threadIndex + 1] - 
            2 * currentPoints[threadIndex] + 
            currentPoints[threadIndex - 1]
        );
        __syncthreads();  // Synchronize after writing nextPoints

        currentPoints[threadIndex] = nextPoints[threadIndex];  // Update currentPoints
        currentTime += dt;  // Increment time

        __syncthreads();  // Synchronize before next iteration
    }
}