#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void xmpS2C_kernel(uint32_t N, uint32_t limbs, uint32_t stride, const uint32_t * in, uint32_t * out) {
    // Calculate thread's global index in x and y dimensions
    uint32_t global_idx_x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t global_idx_y = blockIdx.y * blockDim.y + threadIdx.y;

    // Optimize thread loop by checking boundary conditions upfront
    if(global_idx_x < limbs && global_idx_y < N) {
        // Write coalesced in outer dimension
        out[global_idx_y * limbs + global_idx_x] = in[global_idx_x * stride + global_idx_y];
    }
}