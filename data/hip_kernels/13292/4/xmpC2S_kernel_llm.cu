#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void xmpC2S_kernel(uint32_t N, uint32_t limbs, uint32_t stride, const uint32_t * in, uint32_t * out) {
    // Calculate the unique thread index within the grid
    uint32_t globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t globalIdy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure threads remain within valid range
    if (globalIdx < N && globalIdy < limbs) {
        // Optimize memory access to improve coalescing
        out[globalIdy * stride + globalIdx] = in[globalIdx * limbs + globalIdy];
    }
}