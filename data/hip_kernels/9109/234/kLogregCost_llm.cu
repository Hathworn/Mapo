#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    // Ensure this thread operates within the valid range
    if (tx < numCases) {
        const int label = static_cast<int>(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Calculate correct probability based on maximum probability match
        if (labelp != maxp) {
            correctProbs[tx] = 0.0f;
        } else {
            int numMax = 0;

            // Optimize loop by using a single multiplication outside
            probs += tx;
            for (int i = 0; i < numOut; ++i) {
                numMax += probs[i * numCases] == maxp;
            }
            correctProbs[tx] = 1.0f / static_cast<float>(numMax);
        }
    }
}