#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    const int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate unique thread index
    if (idx == 0) { // Ensure only one thread performs the division
        *accuracy /= N;
    }
}