#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate global thread index for non-overlapping seed sequences
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize CURAND state with calculated sequence number
    hiprand_init(seed, tidx, 0, &state[tidx]);
}