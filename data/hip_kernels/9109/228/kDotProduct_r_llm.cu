#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;

    // Improved loop, avoiding unnecessary reg computation for eidx.
    if (eidx < numElements) {
        for (; eidx < numElements; eidx += DP_BLOCKSIZE * gridDim.x) {
            shmem[threadIdx.x] += a[eidx] * b[eidx];
        }
    }
    __syncthreads();

    // Optimized reduction using loop approach for better maintainability and scalability.
    for (uint offset = DP_BLOCKSIZE / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            shmem[threadIdx.x] += shmem[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Final storage of results
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}