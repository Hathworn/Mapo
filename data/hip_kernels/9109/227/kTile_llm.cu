#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Calculate unique thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Compute total number of threads
    const int numThreads = blockDim.x * gridDim.x;
    // Loop without recalculating tgtWidth * tgtHeight
    const uint numEls = tgtWidth * tgtHeight;
    
    for (uint i = idx; i < numEls; i += numThreads) {
        // Optimize division and modulus with a single operation
        const uint y = i / tgtWidth;
        const uint x = i - y * tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Efficiently copy element from source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}