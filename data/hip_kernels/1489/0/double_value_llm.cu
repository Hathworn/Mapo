#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function with block and thread indexing
__global__ void double_value(double *x, double *y)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    y[idx] = 2.0 * x[idx]; // Perform operation with global index
}