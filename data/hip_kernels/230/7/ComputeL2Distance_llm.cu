#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeL2Distance(float *corrData, int numPts1) {
    // Calculate global point index
    const int p1 = blockIdx.x * blockDim.x + threadIdx.x;
    const int p2 = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check bounds for p1
    if (p1 < numPts1) {
        const int idx = p1 * gridDim.y * blockDim.y + p2;
        
        // Use a conditional operator for conciseness
        corrData[idx] = (corrData[idx] > -1) ? (2 - 2 * corrData[idx]) : FLT_MAX;
    }
}