#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initCURAND(unsigned int seed, hiprandState_t* d_states) {
    // Initialize state with unique sequence per block and thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &d_states[idx]);
}