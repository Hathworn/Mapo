#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__constant__ float d_Threshold[2];
__constant__ float d_Scales[8], d_Factor;
__constant__ float d_EdgeLimit;
__constant__ int d_MaxNumPoints;

__device__ unsigned int d_PointCounter[1];
__constant__ float d_Kernel1[5];
__constant__ float d_Kernel2[12*16];

__global__ void ScaleDown_D(float *d_Result, float *d_Data, int width, int pitch, int height, int newpitch) {
    __shared__ float inrow[SCALEDOWN_W + 4];
    __shared__ float brow[5 * (SCALEDOWN_W / 2)];

    __shared__ int yRead[SCALEDOWN_H + 4];
    __shared__ int yWrite[SCALEDOWN_H + 4];

    const int tx = threadIdx.x;

    // Optimize by calculating dx2 outside the loops
    const int dx2 = SCALEDOWN_W / 2;
    const int txIndices[5] = { tx, tx + dx2, tx + 2 * dx2, tx + 3 * dx2, tx + 4 * dx2 };

    const int xStart = blockIdx.x * SCALEDOWN_W;
    const int yStart = blockIdx.y * SCALEDOWN_H;
    const int xWrite = xStart / 2 + tx;
    int xRead = xStart + tx - 2;
    xRead = (xRead < 0 ? 0 : xRead);
    xRead = (xRead >= width ? width - 1 : xRead);

    const float *k = d_Kernel1;

    if (tx < SCALEDOWN_H + 4) {
        int y = yStart + tx - 1;
        y = (y < 0 ? 0 : y);
        y = (y >= height ? height - 1 : y);

        yRead[tx] = y * pitch;
        yWrite[tx] = (yStart + tx - 4) / 2 * newpitch;
    }

    __syncthreads();

    for (int dy = 0; dy < SCALEDOWN_H + 4; dy += 5) {
        inrow[tx] = d_Data[yRead[dy] + xRead];
        __syncthreads();

        if (tx < dx2) {
            brow[txIndices[0]] = k[0] * (inrow[2 * tx] + inrow[2 * tx + 4]) +
                                 k[1] * (inrow[2 * tx + 1] + inrow[2 * tx + 3]) +
                                 k[2] * inrow[2 * tx + 2];
        }

        __syncthreads();

        if (tx < dx2 && dy >= 4 && !(dy & 1)) {
            d_Result[yWrite[dy] + xWrite] = k[2] * brow[txIndices[2]] +
                                            k[0] * (brow[txIndices[0]] + brow[txIndices[4]]) +
                                            k[1] * (brow[txIndices[1]] + brow[txIndices[3]]);
        }

        // Optimize by reducing redundancy in loop
        for (int i = 1; i < 5 && dy + i < SCALEDOWN_H + 4; i++) {
            inrow[tx] = d_Data[yRead[dy + i] + xRead];
            __syncthreads();
            if (tx < dx2) {
                brow[txIndices[i]] = k[0] * (inrow[2 * tx] + inrow[2 * tx + 4]) +
                                     k[1] * (inrow[2 * tx + 1] + inrow[2 * tx + 3]) +
                                     k[2] * inrow[2 * tx + 2];
            }
            __syncthreads();

            int cond = ((dy + i - 1) >= 0 && ((dy + i - 1) & 1) != (i & 1));
            if (tx < dx2 && cond) {
                d_Result[yWrite[dy + i] + xWrite] = k[2] * brow[txIndices[(i+2)%5]] + 
                                                    k[0] * (brow[txIndices[i]] + brow[txIndices[(i+3)%5]]) +
                                                    k[1] * (brow[txIndices[(i+1)%5]] + brow[txIndices[(i+4)%5]]);
            }
        }
        __syncthreads();
    }
}