#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for dividing accuracy by N
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    if (idx == 0) { // Only one thread performs the division to avoid race condition
        *accuracy /= N;
    }
}