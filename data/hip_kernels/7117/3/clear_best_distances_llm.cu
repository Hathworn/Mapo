#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clear_best_distances(int *best_distances, int rays_number) {
    // Use a grid-stride loop to ensure all threads do useful work
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < rays_number; i += blockDim.x * gridDim.x) {
        best_distances[i] = INT32_MAX;
    }
}