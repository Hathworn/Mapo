#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMaximum(float* __restrict__ mat1, float* __restrict__ mat2, float* __restrict__ target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we do not exceed the bounds
    if (idx < len) {
        target[idx] = fmaxf(mat1[idx], mat2[idx]);
    }
}