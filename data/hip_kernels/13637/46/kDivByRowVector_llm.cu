#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < width * height; i += stride) {
        // Perform division using precomputed indices to minimize calculations
        unsigned int rowIdx = i / width;
        tgtMat[i] = mat[i] / vec[rowIdx];
    }
}