#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kWhere(float* condition_mat, float* if_mat, float* else_mat, float* target, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop with stride equal to total number of threads
    if (idx < len) {
        target[idx] = condition_mat[idx] ? if_mat[idx] : else_mat[idx];
    }
}