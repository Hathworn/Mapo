#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMult(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate unique thread ID and stride for processing
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Optimize by using a stride loop for better memory coalescing
    for (unsigned int i = idx; i < numEls; i += stride) {
        dest[i] = a[i] * b[i];
    }
}