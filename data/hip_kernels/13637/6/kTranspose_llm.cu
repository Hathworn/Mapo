#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTranspose(float *odata, float *idata, int width, int height) {
    __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE+1];

    // Calculate indices outside the boundary check to prevent redundant calculations
    unsigned int xIndex_in = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex_in = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;
    unsigned int index_in = yIndex_in * width + xIndex_in;

    unsigned int xIndex_out = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
    unsigned int yIndex_out = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;
    unsigned int index_out = yIndex_out * height + xIndex_out;

    // Read the matrix tile into shared memory, only if indices are within bounds
    if (xIndex_in < width && yIndex_in < height) {
        block[threadIdx.y][threadIdx.x] = idata[index_in];
    }

    __syncthreads();

    // Write the transposed matrix tile to global memory, only if indices are within bounds
    if (xIndex_out < height && yIndex_out < width) {
        odata[index_out] = block[threadIdx.x][threadIdx.y];
    }
}