#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddScalar(float* a, float alpha, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Loop optimised with a stride variable
    for (unsigned int i = idx; i < numEls; i += stride) {
        dest[i] = a[i] + alpha;
    }
}