#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kEqualsScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Use cache and check boundary condition
    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = mat[i] == val;
    }
}