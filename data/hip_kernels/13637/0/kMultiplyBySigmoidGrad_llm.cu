#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultiplyBySigmoidGrad(float* act, float* target, const unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;  // Use 'stride' instead of 'numThreads' for clarity

    for(unsigned int i = idx; i < len; i += stride) {  // Simplified loop increment variable
        const float activation = act[i];  // Read 'act[i]' once and reuse for performance
        target[i] *= activation * (1.0f - activation);  // Directly update 'target[i]'
    }
}