#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kReciprocal(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (idx < len) {
        target[idx] = 1.f / mat[idx];
    }
}