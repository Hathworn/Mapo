#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSign(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Using loop unrolling for better performance
    for (unsigned int i = idx; i < len; i += stride * 4) {
        if (i < len) target[i] = copysignf(1.0f, mat[i]);
        if (i + stride < len) target[i + stride] = copysignf(1.0f, mat[i + stride]);
        if (i + 2 * stride < len) target[i + 2 * stride] = copysignf(1.0f, mat[i + 2 * stride]);
        if (i + 3 * stride < len) target[i + 3 * stride] = copysignf(1.0f, mat[i + 3 * stride]);
    }
}