#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kEquals(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Optimized loop with stride access pattern
    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = (mat1[i] == mat2[i]) ? 1.0f : 0.0f;  // Ensure target[i] is float
    }
}