#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDivByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index is within bounds
    if (idx < width * height) {
        unsigned int colIndex = idx / height;
        tgtMat[idx] = mat[idx] / vec[colIndex];
    }
}