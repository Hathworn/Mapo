#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kMinimumScalar(float* mat, float val, float* target, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Unroll loop for better performance
    for (unsigned int i = idx; i < len; i += blockDim.x * gridDim.x) {
        target[i] = fminf(mat[i], val);
    }
}