#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetSelectedRows(float* target, float* source, float* indices, int nRowIs, int nCols, int nTargetRows) {
    __shared__ int targetRowIndices[32];
    const int startSourceRowI = blockIdx.x * 32;
    const int tid = threadIdx.x;
    const int localNRowIs = min(32, nRowIs - startSourceRowI);

    // Cooperatively load 32 row indices
    if (tid < localNRowIs) {
        int index = int(indices[startSourceRowI + tid]);
        index = index < 0 ? index + nTargetRows : index; // Normalize negative indices
        targetRowIndices[tid] = (index < 0 || index >= nTargetRows) ? -1 : index; // Validate range
    }
    __syncthreads();

    // Copy 32 rows
    for (int i = 0; i < localNRowIs; i++) {
        const int sourceRowI = startSourceRowI + i;
        const int targetRowI = targetRowIndices[i];
        if (targetRowI != -1) { // Skip invalid target row indices
            for (int colI = tid; colI < nCols; colI += 32) {
                target[targetRowI * nCols + colI] = source[sourceRowI * nCols + colI];
            }
        }
    }
}