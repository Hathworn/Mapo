#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSelectRows(float* source, float* target, float* indices, int nRowIs, int nCols, int nSourceRows) {
    __shared__ int sourceRowIndices[32];
    const int startTargetRowI = blockIdx.x * blockDim.x; // Adjusted for dynamic block size
    const int tid = threadIdx.x;
    const int localNRowIs = min(blockDim.x, nRowIs - startTargetRowI);

    // Load row indices collaboratively
    if (tid < localNRowIs) {
        int idx = int(indices[startTargetRowI + tid]);
        if (idx < 0) idx += nSourceRows; // Adjust negative indices
        sourceRowIndices[tid] = (idx < 0 || idx >= nSourceRows) ? -1 : idx; // Mark invalid indices
    }
    __syncthreads(); 

    // Copy rows with optimized shared memory access
    for (int i = 0; i < localNRowIs; i++) {
        const int targetRowI = startTargetRowI + i;
        const int sourceRowI = sourceRowIndices[i];
        for (int colI = tid; colI < nCols; colI += blockDim.x) {
            target[targetRowI * nCols + colI] = (sourceRowI == -1) ? (1.0 / 0.0 - 1.0 / 0.0) : source[sourceRowI * nCols + colI];
        }
    }
}