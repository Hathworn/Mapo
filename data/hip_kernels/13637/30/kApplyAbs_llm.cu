#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kApplyAbs(float* mat, float* target, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Utilize linear-thread cache for reduced memory access latency
    if (idx < len) {
        target[idx] = fabsf(mat[idx]);
    }
}