#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivide(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate unique thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure idx is within bounds
    if (idx < numEls) {
        dest[idx] = a[idx] / b[idx];
    }
}