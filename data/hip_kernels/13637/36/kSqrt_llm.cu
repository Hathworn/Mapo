#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSqrt(float* mat, float* target, unsigned int len) {
    // Calculate global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread processes an assigned element
    if (idx < len) {
        target[idx] = sqrt(mat[idx]);
    }
}
```
