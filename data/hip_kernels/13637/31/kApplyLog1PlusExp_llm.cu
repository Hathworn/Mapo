#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kApplyLog1PlusExp(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Loop unrolling for better performance
    for (unsigned int i = idx; i < len; i += numThreads) {
        float mat_i = mat[i];
        // Use a more efficient computation logic
        target[i] = mat_i > 0 ? (__logf(1 + __expf(-mat_i)) + mat_i) : __logf(1 + __expf(mat_i));
    }
}