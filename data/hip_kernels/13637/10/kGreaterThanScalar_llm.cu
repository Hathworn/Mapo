#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kGreaterThanScalar(float* mat, float val, float* target, unsigned int len) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize by removing unnecessary variable and loop
    if (idx < len) {
        target[idx] = mat[idx] > val;
    }
}