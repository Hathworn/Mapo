#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultScalar(float* mat, float alpha, float* dest, unsigned int len) {
    // Calculate unique thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure idx within bounds to avoid redundant computation
    if (idx < len) {
        dest[idx] = alpha * mat[idx];
    }
}