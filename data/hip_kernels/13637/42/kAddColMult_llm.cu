#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddColMult(float* mat, float* vec, float* tgtMat, float mult, unsigned int width, unsigned int height) {
    // Calculate global index
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Stride loop with multiple elements per thread
    unsigned int stride = blockDim.x * gridDim.x;
    for (unsigned int i = idx; i < width * height; i += stride) {
        tgtMat[i] = mat[i] + mult * vec[i % height];
    }
}