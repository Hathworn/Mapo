#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Optimize by calculating thread index only once and using shared memory for the vector
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    __shared__ float sharedVec[1024]; // Assuming a block size that does not exceed 1024

    for (unsigned int i = idx; i < width * height; i += stride) {
        if (threadIdx.x < height) {
            sharedVec[threadIdx.x] = vec[threadIdx.x];
        }
        __syncthreads();
        tgtMat[i] = mat[i] * sharedVec[i % height];
        __syncthreads();
    }
}