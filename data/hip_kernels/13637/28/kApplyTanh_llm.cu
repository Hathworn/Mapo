#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kApplyTanh(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Loop unrolling for better performance
    for (unsigned int i = idx; i < len; i += numThreads * 4) {
        if (i < len) {
            float mat_i = mat[i];
            float exp2x = __expf(2 * mat_i);
            target[i] = 1 - 2 / (exp2x + 1);
        }
        if (i + numThreads < len) {
            float mat_i1 = mat[i + numThreads];
            float exp2x1 = __expf(2 * mat_i1);
            target[i + numThreads] = 1 - 2 / (exp2x1 + 1);
        }
        if (i + 2 * numThreads < len) {
            float mat_i2 = mat[i + 2 * numThreads];
            float exp2x2 = __expf(2 * mat_i2);
            target[i + 2 * numThreads] = 1 - 2 / (exp2x2 + 1);
        }
        if (i + 3 * numThreads < len) {
            float mat_i3 = mat[i + 3 * numThreads];
            float exp2x3 = __expf(2 * mat_i3);
            target[i + 3 * numThreads] = 1 - 2 / (exp2x3 + 1);
        }
    }
}