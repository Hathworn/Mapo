#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMinimum(float* mat1, float* mat2, float* target, unsigned int len) {
    // Calculate the global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the index is within bounds
    if (idx < len) {
        target[idx] = fminf(mat1[idx], mat2[idx]);
    }
}