#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLog(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) { // Process only if index is within bounds
        target[idx] = __logf(mat[idx]);
    }
}