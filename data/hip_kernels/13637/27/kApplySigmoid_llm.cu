#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplySigmoid(float* mat, float* target, unsigned int len) {
    // Use grid-stride loop to ensure all elements are processed
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < len; i += blockDim.x * gridDim.x) {
        target[i] = 1 / (1 + __expf(-mat[i]));
    }
}