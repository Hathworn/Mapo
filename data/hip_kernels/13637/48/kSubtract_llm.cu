#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSubtract(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate unique thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds before computation
    if (idx < numEls) {
        dest[idx] = a[idx] - b[idx];
    }
}