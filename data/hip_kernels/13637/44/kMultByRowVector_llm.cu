#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMultByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate indices for the current thread
    const unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure the thread accesses a valid element
    if (col < width && row < height) {
        unsigned int i = row * width + col;
        tgtMat[i] = mat[i] * vec[row];
    }
}