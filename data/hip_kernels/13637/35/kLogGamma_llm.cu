#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogGamma(float* mat, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Use loop unrolling to increase instruction-level parallelism
    for (unsigned int i = idx; i < len; i += numThreads * 4) {
        if (i < len) target[i] = lgammaf(mat[i]);
        if (i + numThreads < len) target[i + numThreads] = lgammaf(mat[i + numThreads]);
        if (i + 2 * numThreads < len) target[i + 2 * numThreads] = lgammaf(mat[i + 2 * numThreads]);
        if (i + 3 * numThreads < len) target[i + 3 * numThreads] = lgammaf(mat[i + 3 * numThreads]);
    }
}