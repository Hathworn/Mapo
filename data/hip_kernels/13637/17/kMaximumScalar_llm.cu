#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kMaximumScalar(float* mat, float val, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int numThreads = blockDim.x * gridDim.x;

    // Unroll the loop to increase instruction throughput
    for (unsigned int i = idx; i < len; i += numThreads * 4) {
        if (i < len) target[i] = fmaxf(mat[i], val);
        if (i + numThreads < len) target[i + numThreads] = fmaxf(mat[i + numThreads], val);
        if (i + 2 * numThreads < len) target[i + 2 * numThreads] = fmaxf(mat[i + 2 * numThreads], val);
        if (i + 3 * numThreads < len) target[i + 3 * numThreads] = fmaxf(mat[i + 3 * numThreads], val);
    }
}