#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAddColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    const unsigned int column = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure the thread is within matrix bounds
    if (column < width && row < height) {
        unsigned int index = row * width + column;
        // Optimize by accessing memory in a coalesced manner
        tgtMat[index] = mat[index] + vec[row];
    }
}
```
