#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLessThanScalar(float* mat, float val, float* target, unsigned int len) {
    // Calculate global thread index
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    // Utilize coalesced memory access and avoid bank conflicts
    for (unsigned int i = index; i < len; i += stride) {
        target[i] = mat[i] < val ? 1.0f : 0.0f;
    }
}