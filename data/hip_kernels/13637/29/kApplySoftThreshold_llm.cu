#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kApplySoftThreshold(float* mat, float alpha, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    // Optimize to use threadIdx.x for better memory coalescing
    for (unsigned int i = idx; i < len; i += stride) {
        float f = mat[i];
        target[i] = (f > alpha) ? (f - alpha) : ((f < -alpha) ? (f + alpha) : 0.0f);
    }
}