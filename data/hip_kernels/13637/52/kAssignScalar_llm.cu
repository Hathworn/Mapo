#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAssignScalar(float* dest, float alpha, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x; // Use a more descriptive name for stride

    // Use a conditional check to eliminate unnecessary iterations
    if (idx < len) {
        for (unsigned int i = idx; i < len; i += stride) {
            dest[i] = alpha;
        }
    }
}