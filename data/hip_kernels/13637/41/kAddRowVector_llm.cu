#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAddRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
    // Calculate global index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Calculate row and column based on global index
    unsigned int row = idx / width;
    unsigned int col = idx % width;

    // Ensure the index is within bounds
    if (row < height && col < width) {
        // Update the target matrix with the element-wise sum using row vector
        tgtMat[row * width + col] = mat[row * width + col] + vec[col];
    }
}