#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kPowMatrix(float* mat, float* pow, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure valid memory access
    if (idx < len) {
        target[idx] = powf(mat[idx], pow[idx]);
    }
}
```
