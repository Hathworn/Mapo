#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kGamma(float* mat, float* target, unsigned int len) {
    // Optimize: pre-calculate grid stride outside loop
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;

    for (unsigned int i = idx; i < len; i += stride) {
        target[i] = tgammaf(mat[i]);
    }
}