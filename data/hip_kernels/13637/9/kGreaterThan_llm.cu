#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kGreaterThan(float* mat1, float* mat2, float* target, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < len) {
        target[idx] = mat1[idx] > mat2[idx];  // Use thread index directly for improved memory access pattern
    }
}