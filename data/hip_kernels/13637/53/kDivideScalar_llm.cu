#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDivideScalar(float* mat, float alpha, float* dest, unsigned int len) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if index is within bounds to avoid unnecessary calculations
    if (idx < len) {
        // Perform division only for valid indices
        dest[idx] = mat[idx] / alpha;
    }
}