#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop for efficient memory access and parallelism
    for (unsigned int i = idx; i < numEls; i += blockDim.x * gridDim.x) {
        dest[i] = a[i] + b[i]; // Perform element-wise addition
    }
}