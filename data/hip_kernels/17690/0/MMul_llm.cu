#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif
float a[1024][1024], b[1024][1024], c[1024][1024];

// Optimized matrix multiplication kernel
__global__ void MMul(float *m, float *d, float *p, int n) {
    int r = blockIdx.y * blockDim.y + threadIdx.y; // row
    int c = blockIdx.x * blockDim.x + threadIdx.x; // column
    if (r < n && c < n) { // Check bounds
        float p_sum = 0;
        for (int i = 0; i < n; i++) {
            p_sum += m[r * n + i] * d[i * n + c]; // Correct accumulation operator
        }
        p[r * n + c] = p_sum;
    }
}