#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_copy_rc_cu_z(const double *din, double *zout, const int n) {
    int igpt = blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x; // Optimize index calculation

    if (igpt < n) {
        double value = din[igpt]; // Cache input value
        zout[2 * igpt] = value;
        zout[2 * igpt + 1] = 0.0; // Make zero assignment cleaner
    }
}