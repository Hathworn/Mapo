#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_scatter_cu_z(double *c, const double *pwcc, const double scale, const int ngpts, const int nmaps, const int *ghatmap) {

    // Use variables to avoid redundant calculations
    int igpt = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

    if (igpt < ngpts) {
        int index = 2 * igpt;
        int mapIndex = 2 * ghatmap[igpt];

        // Combined loading, scaling, and writing to optimize memory operations
        double scaledReal = scale * pwcc[index];
        double scaledImag = scale * pwcc[index + 1];

        c[mapIndex] = scaledReal;
        c[mapIndex + 1] = scaledImag;

        if (nmaps == 2) {
            int mapIndex_nmaps2 = 2 * ghatmap[igpt + ngpts];
            c[mapIndex_nmaps2] = scaledReal;
            c[mapIndex_nmaps2 + 1] = -scaledImag; // Remove unneeded multiplication
        }
    }
}