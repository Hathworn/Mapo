#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_copy_cr_cu_z(const double *zin, double *dout, const int n) {
    // Calculate global thread index
    int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Check if index is within bounds
    if (igpt < n) {
        // Copy data from input to output
        dout[igpt] = zin[2 * igpt];
    }
}
