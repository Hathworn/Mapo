#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pw_gather_cu_z(double *pwcc, const double *c, const double scale, const int ngpts, const int *ghatmap) {

    const int igpt = blockIdx.x * blockDim.x + threadIdx.x;

    if (igpt < ngpts) {
        int map_idx = ghatmap[igpt];
        double scale_val = scale;

        // Avoid redundant memory access by storing values in registers
        double real_part = scale_val * c[2 * map_idx];
        double imag_part = scale_val * c[2 * map_idx + 1];

        // Write results to the output
        pwcc[2 * igpt] = real_part;
        pwcc[2 * igpt + 1] = imag_part;
    }
}
```
