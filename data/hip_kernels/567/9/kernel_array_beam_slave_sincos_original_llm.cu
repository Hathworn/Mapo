#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_array_beam_slave_sincos_optimized(int N, float r1, float r2, float r3, float *x, float *y, float *z, float *sum, int blockDim_2) {
    unsigned int n = threadIdx.x;
    __shared__ float tmpsum[1024]; // Adjust shared memory size to a common block size power of 2

    if (n < N) {
        // Precalculate often used expression to reduce repetitive computation
        float angle = r1 * __ldg(&x[n]) + r2 * __ldg(&y[n]) + r3 * __ldg(&z[n]);
        float ss, cc;
        sincosf(angle, &ss, &cc);
        tmpsum[2 * n] = ss;
        tmpsum[2 * n + 1] = cc;
    }
    __syncthreads();

    // Optimize reduction by ensuring memory coalescing
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (n < stride && n + stride < blockDim_2 / 2) {
            tmpsum[2 * n] += tmpsum[2 * (n + stride)];
            tmpsum[2 * n + 1] += tmpsum[2 * (n + stride) + 1];
        }
        __syncthreads();
    }

    // Store the result from thread 0
    if (threadIdx.x == 0) {
        sum[0] = tmpsum[0];
        sum[1] = tmpsum[1];
    }
}