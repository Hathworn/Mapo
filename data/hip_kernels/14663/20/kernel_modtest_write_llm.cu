#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    unsigned int i;
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);

    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Unroll loop for better performance
    for (i = offset; i < BLOCKSIZE / sizeof(unsigned int); i += MOD_SZ) {
        ptr[i] = p1;
    }

    // Optimize: Combine two loops and use threadIdx for parallel write
    for (i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        if (i % MOD_SZ != offset) {
            ptr[i] = p2;
        }
    }

    return;
}