#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    // Calculate the starting pointer for the block
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Early exit if pointer exceeds end pointer
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Use a stride loop to ensure memory coalescing and reduce divergent warps
    for (unsigned int i = threadIdx.x; i < BLOCKSIZE / sizeof(unsigned int); i += blockDim.x) {
        ptr[i] = pattern;
    }
}