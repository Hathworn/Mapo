#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    // Calculate the starting pointer for this block
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    
    // Return early if the starting pointer is beyond end_ptr
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Initialize variables
    unsigned int k = offset;
    unsigned int pat = pattern;
    
    // Calculate number of elements to process in this kernel
    unsigned int num_elements = BLOCKSIZE / sizeof(unsigned int);
    
    // Loop over each element in the block efficiently
    for (unsigned int i = threadIdx.x; i < num_elements; i += blockDim.x){
        ptr[i] = pat; // Assign pattern to current position
        k++; // Increment k
        if (k >= 32) {
            k = 0; // Reset k if it exceeds 31
            pat = lb; // Reset pattern to new pattern when k exceeds 31
        } else {
            pat = (pat << 1) | sval; // Shift pattern left and set sval
        }
    }
}