#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);

    // Check if pointer exceeds end pointer early
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Unrolled loop for better performance
    #pragma unroll
    for (unsigned int i = 0; i < half_count; i++) {
        ptr_mid[i] = ptr[i];
    }

    // Unrolled loop for better performance
    #pragma unroll
    for (unsigned int i = 0; i < half_count - 8; i++) {
        ptr[i + 8] = ptr_mid[i];
    }

    // Unrolled loop for better performance
    #pragma unroll
    for (unsigned int i = 0; i < 8; i++) {
        ptr[i] = ptr_mid[half_count - 8 + i];
    }
}