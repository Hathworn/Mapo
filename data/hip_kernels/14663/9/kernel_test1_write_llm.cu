#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    // Calculate the base address of the current block
    unsigned long* ptr = (unsigned long*)(_ptr + blockIdx.x * BLOCKSIZE);

    // Check if the calculated pointer exceeds the end pointer
    if (ptr >= (unsigned long*)end_ptr) {
        return;
    }

    // Calculate the start index for the current thread in the block
    unsigned int index = threadIdx.x;

    // Unroll the loop to improve performance
    #pragma unroll
    for (unsigned int i = index; i < BLOCKSIZE / sizeof(unsigned long); i += blockDim.x) {
        // Write to memory with the address of the pointer
        ptr[i] = (unsigned long)&ptr[i];
    }
}