#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    // Calculate the global index for the threads
    unsigned int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int* ptr = (unsigned int*)_ptr;

    // Calculate the number of unsigned ints to process per block
    unsigned int numIntsPerBlock = BLOCKSIZE / sizeof(unsigned int);

    // Use the global index to determine the offset within ptr
    unsigned int* write_ptr = ptr + numIntsPerBlock * blockIdx.x;

    // Avoid out-of-bound writes
    if ((char*)(write_ptr + threadIdx.x) >= end_ptr) {
        return;
    }

    unsigned int* start_ptr = (unsigned int*)_start_ptr;

    // Only the thread within block size should write
    if (threadIdx.x < numIntsPerBlock) {
        write_ptr[threadIdx.x] = start_ptr[threadIdx.x]; // Copy data from start_ptr with each thread writing unique data 
    }
}
```
