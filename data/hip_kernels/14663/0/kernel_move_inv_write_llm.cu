#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Optimize memory access using threadIdx.x for parallelism
    unsigned int idx = threadIdx.x;
    unsigned int stride = BLOCKSIZE / sizeof(unsigned int);
    for (unsigned int i = idx; i < stride; i += blockDim.x) {
        ptr[i] = pattern;
    }
}