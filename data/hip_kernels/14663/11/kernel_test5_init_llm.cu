#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_init(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check if kernel operation is out of bounds
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int p1 = 1;
    unsigned int p2; // Define p2 once

    // Unroll loop manually for optimization; avoid unnecessary calculations
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i += 16) {
        p2 = ~p1; // Compute p2 outside of storing to avoid repetition

        #pragma unroll // Suggest compiler to unroll loop for better performance
        for (int j = 0; j < 16; j += 4) { 
            ptr[i + j]     = (j % 8 < 4) ? p1 : p2;
            ptr[i + j + 1] = (j % 8 < 4) ? p1 : p2;
            ptr[i + j + 2] = (j % 8 < 4) ? p2 : p1;
            ptr[i + j + 3] = (j % 8 < 4) ? p2 : p1;
        }

        p1 = p1 << 1; // Increment p1
        if (p1 == 0) {
            p1 = 1; // Reset p1
        }
    }
}