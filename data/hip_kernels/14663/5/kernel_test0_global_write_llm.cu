#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    // Using shared memory to reduce global memory access
    __shared__ unsigned int shared_pattern;
    __shared__ unsigned long shared_mask;
    
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int* orig_ptr = ptr;

    // Initialize shared variables
    if (threadIdx.x == 0) {
        shared_pattern = 1;
        shared_mask = 4;
    }
    __syncthreads();

    *ptr = shared_pattern;

    while(ptr < end_ptr){
        ptr = (unsigned int*) (((unsigned long)orig_ptr) | shared_mask);
        if (ptr == orig_ptr){
            shared_mask = shared_mask << 1;
            continue;
        }
        if (ptr >= end_ptr){
            break;
        }

        *ptr = shared_pattern;

        shared_pattern = shared_pattern << 1;
        shared_mask = shared_mask << 1;
    }
    return;
}