#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    // Calculate the starting pointer for the current block
    unsigned int* orig_ptr = reinterpret_cast<unsigned int*>(_ptr) + blockIdx.x * (BLOCKSIZE / sizeof(unsigned int));
    unsigned int* ptr = orig_ptr;

    // Early exit if the block's starting point exceeds the end pointer
    if (ptr >= reinterpret_cast<unsigned int*>(end_ptr)) {
        return;
    }

    // Define block_end as the end of the block size
    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);

    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Write the initial pattern to the start of the block
    *ptr = pattern;

    // Optimize loop to minimize pointer arithmetic inside the loop
    while (true) {
        // Calculate the next pointer with mask
        ptr = reinterpret_cast<unsigned int*>(reinterpret_cast<unsigned long>(orig_ptr) | mask);
        
        if (ptr >= block_end) {
            break;
        }

        // Only write to memory if ptr is a different pointer than orig_ptr
        *ptr = pattern;
        
        // Update pattern and mask for the next iteration
        if (ptr == orig_ptr) {
            mask <<= 1;
            continue;
        }
        
        pattern <<= 1;
        mask <<= 1;
    }
}