#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define width 100
#define height 72

unsigned char results[width * height];
unsigned char image[] = {/*...*/};

__global__ void detect_edges(unsigned char *input, unsigned char *output) {
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Improved indexing
    
    if (i >= width * height) return; // Ensure within bounds

    int y = i / width;
    int x = i % width; // Simplified calculation for x
    
    // Check borders
    if (x == 0 || y == 0 || x == width - 1 || y == height - 1) {
        output[i] = 0;
    } else {
        // Calculate adjacent indices
        int b = (y + 1) * width + x; // below
        int d = i - 1;               // left
        int f = i + 1;               // right
        int h = (y - 1) * width + x; // above

        int r = input[i] * 4 - input[b] - input[d] - input[f] - input[h];

        // Output result based on calculation
        output[i] = (r >= 0) ? 255 : 0;
    }
}