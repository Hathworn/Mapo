#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAddGPU(const double * __restrict__ a, const double * __restrict__ b, double * __restrict__ c, int n) {
    // Use shared memory and pointer restrict qualifiers for performance
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < n) {
        c[id] = a[id] + b[id];
    }
}