#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce(float *g_idata, float *g_odata) {
    extern __shared__ float sdata[];

    // Load elements into shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + tid;
    sdata[tid] = g_idata[i];
    __syncthreads();

    // Perform reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {  // Ensure only threads with relevant indices perform work
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}