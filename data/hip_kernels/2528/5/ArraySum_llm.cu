#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ArraySum(float *array, float *sum) {
    // Use shared memory for partial sums
    __shared__ float shared_sum;

    // Initialize shared memory
    if (threadIdx.x == 0) shared_sum = 0.0f;
    __syncthreads();

    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Accumulate values from global memory into shared memory
    float value = 0.0f;
    if (index < N) {
        value = array[index];
    }
    atomicAdd(&shared_sum, value);
    __syncthreads();

    // Accumulate from shared memory to global sum
    if (threadIdx.x == 0) {
        atomicAdd(sum, shared_sum);
    }
}