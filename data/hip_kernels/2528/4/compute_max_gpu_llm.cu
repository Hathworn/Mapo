#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_max_gpu(float *device_input, float *device_output) {
    extern __shared__ float sm[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    sm[tid] = device_input[i];
    __syncthreads();

    // Use warp reduction for better efficiency
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sm[tid] = max(sm[tid], sm[tid + s]);
        }
        __syncthreads();
    }

    // Write the result of this block to global memory
    if (tid == 0) device_output[blockIdx.x] = sm[0];
}