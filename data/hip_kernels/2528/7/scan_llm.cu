#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan(float *input, float *output, int len) {
    __shared__ float data[BLOCK_SIZE];

    // Load elements into shared memory.
    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;
    if (index < len) {
        data[tid] = input[index];
    } else {
        data[tid] = 0.0f; // Initialize out-of-bound threads to zero
    }
    __syncthreads();

    // Perform exclusive scan within the block.
    for (int offset = 1; offset < blockDim.x; offset *= 2) {
        float temp = 0.0f;
        if (tid >= offset) {
            temp = data[tid - offset];
        }
        __syncthreads();
        data[tid] += temp;
        __syncthreads();
    }

    // Write results to output array.
    if (index < len) {
        output[index] = data[tid];
    }
}