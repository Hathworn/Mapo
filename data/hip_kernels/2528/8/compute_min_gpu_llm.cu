#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_min_gpu(float *device_input, float *device_output) {
    extern __shared__ float sm[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input into shared memory
    sm[tid] = device_input[i];
    __syncthreads();

    // Efficient reduction using shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sm[tid] = min(sm[tid], sm[tid + s]);
        }
        __syncthreads();
    }

    // Write result of this block to global memory
    if (tid == 0) device_output[blockIdx.x] = sm[0];
}