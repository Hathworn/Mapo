#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(float *A, float *B, float *C, int N) {
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure within bounds
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}
```
