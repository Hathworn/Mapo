#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void VecAdd(float *A, float *B, float *C){
int i = threadIdx.x;
C[i] = A[i] + B[i];
}