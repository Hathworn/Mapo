#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void revisedArraySum(float *array, float *sum) {
    __shared__ float partialSum[256];
    int t = threadIdx.x;
    
    // Initialize the shared memory
    partialSum[t] = array[t];
    __syncthreads();

    // Perform iterative reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (t < stride) {
            partialSum[t] += partialSum[t + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (t == 0) {
        sum[0] = partialSum[0];
    }
}