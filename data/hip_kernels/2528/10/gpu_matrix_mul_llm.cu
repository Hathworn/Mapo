#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_matrix_mul(int *a, int *b, int *c) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    // Check if within matrix bounds
    if (col < N && row < N) {
        // Use shared memory for better memory access patterns
        __shared__ int shared_a[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ int shared_b[BLOCK_SIZE][BLOCK_SIZE];

        for (int i = 0; i < N/BLOCK_SIZE; i++) {
            // Load matrices into shared memory
            shared_a[threadIdx.y][threadIdx.x] = a[row * N + i * BLOCK_SIZE + threadIdx.x];
            shared_b[threadIdx.y][threadIdx.x] = b[(i * BLOCK_SIZE + threadIdx.y) * N + col];
            __syncthreads();

            // Perform partial multiplication
            for (int j = 0; j < BLOCK_SIZE; j++) {
                sum += shared_a[threadIdx.y][j] * shared_b[j][threadIdx.x];
            }
            __syncthreads();
        }
        // Write back the accumulated sum to the result matrix
        c[row * N + col] = sum;
    }
}