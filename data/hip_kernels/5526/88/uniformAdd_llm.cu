#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    __shared__ float uni;

    // Load uniform value into shared memory by the first thread of the block
    if (threadIdx.x == 0)
        uni = uniforms[blockIdx.x + blockOffset];

    // Compute global address using a more efficient calculation
    unsigned int address = blockIdx.x * (blockDim.x << 1) + baseIndex + threadIdx.x;

    // Ensure uniform value is loaded before proceeding
    __syncthreads();

    // Apply uniform addition if within bounds
    if (address < n) 
        g_data[address] += uni;

    // Apply second addition if within bounds
    if (address + blockDim.x < n) 
        g_data[address + blockDim.x] += uni;
}