#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f; // Use register for intermediate sum

    // Loop unrolling for better performance
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        sum += a[eidx] * b[eidx];
    }

    shmem[threadIdx.x] = sum; // Store in shared memory
    __syncthreads();

    // Optimize reduction with fewer __syncthreads
    if (threadIdx.x < 128) {
        shmem[threadIdx.x] += shmem[threadIdx.x + 128];
    }
    __syncthreads();

    if (threadIdx.x < 64) {
        shmem[threadIdx.x] += shmem[threadIdx.x + 64];
    }
    __syncthreads();

    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    // Use first thread to store the result
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}