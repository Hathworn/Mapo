#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate unique index for each thread
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    // Initialize cuRAND state with optimized launch configuration
    hiprand_init(seed, tidx, 0, &state[tidx]);
}