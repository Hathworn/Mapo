#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    // Efficient reduction using warp-level operations
    float tmp = 0.0f;
    for (int i = idx; i < dim; i += blockDim.x) {
        tmp += dY[i] * Y[i];
    }
    reduction_buffer[idx] = tmp;
    __syncthreads();
    
    // Warp-level reduction
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
       tmp += __shfl_down_sync(0xFFFFFFFF, tmp, offset);
    }
    
    if (idx % warpSize == 0) {
        atomicAdd(&reduction_buffer[0], tmp);
    }
    __syncthreads();
    
    // Compute gradient with improved memory access
    tmp = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - tmp);
    }
}