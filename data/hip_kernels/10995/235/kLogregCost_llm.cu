#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = __float2int_rd(labels[tx]); // Use fast conversion
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Early exit if label probability is not max
        if (labelp != maxp) {
            correctProbs[tx] = 0;
        } else {
            int numMax = 0;
            // Use shared memory to minimize global memory accesses
            __shared__ int sharedNumMax[LOGREG_ERR_THREADS_X];
            sharedNumMax[threadIdx.x] = 0; // Initialize shared memory location

            for (int i = threadIdx.x; i < numOut; i += LOGREG_ERR_THREADS_X) {
                if (probs[i * numCases + tx] == maxp) {
                    atomicAdd(&sharedNumMax[threadIdx.x], 1);
                }
            }
            __syncthreads(); // Ensure all threads have completed their additions

            if (threadIdx.x == 0) { // Single thread final aggregation
                numMax = 0;
                for (int i = 0; i < LOGREG_ERR_THREADS_X; ++i) {
                    numMax += sharedNumMax[i];
                }
                correctProbs[tx] = 1.0f / float(numMax);
            }
        }
    }
}