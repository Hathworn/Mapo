#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.y * blockDim.x) + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void complexAbsSum(double2 *in1, double2 *in2, double2 *in3, double *out){
    int gid = getGid3d3d();

    // Use shared memory to reduce global memory accesses
    __shared__ double2 sharedIn1[BLOCK_SIZE];
    __shared__ double2 sharedIn2[BLOCK_SIZE];
    __shared__ double2 sharedIn3[BLOCK_SIZE];

    // Load data into shared memory
    sharedIn1[threadIdx.x] = in1[gid];
    sharedIn2[threadIdx.x] = in2[gid];
    sharedIn3[threadIdx.x] = in3[gid];
    __syncthreads();

    // Perform computation using shared memory
    double2 temp;
    temp.x = sharedIn1[threadIdx.x].x + sharedIn2[threadIdx.x].x + sharedIn3[threadIdx.x].x;
    temp.y = sharedIn1[threadIdx.x].y + sharedIn2[threadIdx.x].y + sharedIn3[threadIdx.x].y;
    out[gid] = sqrt(temp.x * temp.x + temp.y * temp.y);
}