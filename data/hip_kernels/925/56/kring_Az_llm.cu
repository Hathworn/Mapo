#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y))
                 + threadIdx.x;
    return threadId;
}

__global__ void kring_Az(double *x, double *y, double *z, double xMax, double yMax, double zMax, double omegaX, double omegaY, double omegaZ, double omega, double fudge, double *A) {
    int gid = getGid3d3d();
    int xid = blockDim.x * blockIdx.x + threadIdx.x;
    int yid = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Optimized by reusing common subexpression
    double x_val = x[xid];
    double y_val = y[yid];
    double rad = sqrt(x_val * x_val + y_val * y_val);
    
    // Reduced operations by pre-computing constants
    double exp_coeff = omega * 0.01;
    double rad_coeff = -1.0 / (0.0001 * xMax);
    
    A[gid] = exp_coeff * exp(rad * rad * rad_coeff);
}