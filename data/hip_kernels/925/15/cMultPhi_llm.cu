#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + threadIdx.z * (blockDim.x * blockDim.y)
                 + threadIdx.y * blockDim.x + threadIdx.x;
    return threadId;
}

__global__ void cMultPhi(double2* in1, double* in2, double2* out) {
    unsigned int gid = getGid3d3d();
    double phase = in2[gid];  // Pre-compute phase
    double cosPhase = cos(phase);  // Pre-compute cosine
    double sinPhase = sin(phase);  // Pre-compute sine

    double2 input = in1[gid];  // Read input once
    double2 result;
    
    // Use pre-computed values and minimize arithmetic operations
    result.x = cosPhase * input.x - sinPhase * input.y;
    result.y = sinPhase * input.x + cosPhase * input.y;
    
    out[gid] = result;  // Write result
}