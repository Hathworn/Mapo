#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double2 mult(double2 a, double2 b){
    return {a.x*b.x - a.y*b.y, a.x*b.y + a.y*b.x};
}

__device__ double2 mult(double2 a, double b){
    return {a.x*b, a.y*b};
}

__global__ void mult_test(double2 *a, double2 *b, double2 *c){
    // Optimize global memory access by using shared memory and threadIdx for parallel processing.
    __shared__ double2 s_a, s_b;

    if (threadIdx.x == 0) {
        s_a = a[0];
        s_b = b[0];
    }
    __syncthreads();

    if (threadIdx.x == 0) {
        c[0] = mult(s_a, s_b);
    }
}