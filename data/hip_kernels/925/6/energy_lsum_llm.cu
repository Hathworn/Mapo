#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void energy_lsum(double *in1, double2 *in2, double *out){
    int gid = blockIdx.x * blockDim.x * blockDim.y * blockDim.z
            + blockIdx.y * gridDim.x * blockDim.x * blockDim.y * blockDim.z
            + blockIdx.z * gridDim.x * gridDim.y * blockDim.x * blockDim.y * blockDim.z
            + threadIdx.z * blockDim.x * blockDim.y
            + threadIdx.y * blockDim.x
            + threadIdx.x;
    
    // Simplified gid calculation; directly added inputs to output
    if (gid < gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z) {
        out[gid] = in1[gid] + in2[gid].x;
    }
}