#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                   + (threadIdx.y * blockDim.x)
                   + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void simple_K(double *xp, double *yp, double *zp, double mass, double *K) {
    unsigned int gid = getGid3d3d();

    // Use better indexing to avoid multiple global memory accesses
    double x = xp[blockDim.x * blockIdx.x + threadIdx.x];
    double y = yp[blockDim.y * blockIdx.y + threadIdx.y];
    double z = zp[blockDim.z * blockIdx.z + threadIdx.z];

    // Simplify computation
    K[gid] = (HBAR * HBAR / (2 * mass)) * (x * x + y * y + z * z);
}