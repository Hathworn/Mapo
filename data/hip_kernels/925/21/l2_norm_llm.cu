#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
        + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
        + (threadIdx.y * blockDim.x)
        + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void l2_norm(double2 *in1, double2 *in2, double2 *in3, double *out) {
    int gid = getGid3d3d();

    // Load values into registers to reduce global memory access latency
    double2 val1 = in1[gid];
    double2 val2 = in2[gid];
    double2 val3 = in3[gid];

    // Calculate the sum of squares to improve readability
    double sum_squares = val1.x * val1.x + val1.y * val1.y
        + val2.x * val2.x + val2.y * val2.y
        + val3.x * val3.x + val3.y * val3.y;

    // Compute square root and store in output array
    out[gid] = sqrt(sum_squares);
}