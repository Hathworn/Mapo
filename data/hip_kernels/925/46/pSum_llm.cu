#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.y * blockDim.x) 
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void pSum(double* __restrict__ in1, double* __restrict__ output, int pass) {
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.y * gridDim.x + blockIdx.x; // Optimized bid calculation
    extern __shared__ double sdata2[];

    // Pre-load shared memory with input data
    sdata2[tid] = in1[getGid3d3d()];
    __syncthreads();

    for(int i = blockDim.x >> 1; i > 0; i >>= 1) {
        if(tid < i) { // Reduced unnecessary loads and checks
            sdata2[tid] += sdata2[tid + i];
        }
        __syncthreads();
    }

    if(tid == 0) {
        output[bid] = sdata2[0];
    }
}