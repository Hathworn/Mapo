#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void scalarDiv(double2* in, double factor, double2* out){
    // Improved memory access coalescing
    unsigned int gid = getGid3d3d();
    double2 input = in[gid];
    double invFactor = 1.0 / factor;  // Use reciprocal for division

    double2 result;
    result.x = input.x * invFactor;
    result.y = input.y * invFactor;

    out[gid] = result;
}