#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

// Optimized global kernel function
__global__ void scalarDiv(double* in, double factor, double* out){
    unsigned int gid = getGid3d3d();
    
    // Perform division and store result
    out[gid] = in[gid] / factor;
}