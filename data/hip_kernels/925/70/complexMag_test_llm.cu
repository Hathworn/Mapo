#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Calculate the magnitude of a complex number
__device__ double complexMagnitude(double2 in) {
    return sqrt(in.x*in.x + in.y*in.y);
}

// Using grid-stride loop for improved parallelism and efficiency
__global__ void complexMag_test(double2 *in, double *out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    for (int i = idx; i < N; i += stride) {
        out[i] = complexMagnitude(in[i]);
    }
}