#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define THREADS_PER_BLOCK_X blockDim.x
#define THREADS_PER_BLOCK_Y blockDim.y
#define THREADS_PER_BLOCK_Z blockDim.z

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (THREADS_PER_BLOCK_X * THREADS_PER_BLOCK_Y * THREADS_PER_BLOCK_Z)
                 + (threadIdx.z * THREADS_PER_BLOCK_Y * THREADS_PER_BLOCK_X)
                 + (threadIdx.y * THREADS_PER_BLOCK_X) + threadIdx.x;
    return threadId;
}

__global__ void scalarDiv_wfcNorm(double2* in, double dr, double* pSum, double2* out){
    unsigned int gid = getGid3d3d(); // Compute unique thread index

    if (gid < gridDim.x * gridDim.y * gridDim.z * THREADS_PER_BLOCK_X * THREADS_PER_BLOCK_Y * THREADS_PER_BLOCK_Z) {
        double2 result;
        double norm = rsqrt(pSum[0] * dr); // Use rsqrt for performance

        result.x = in[gid].x * norm; // Optimize division to multiplication
        result.y = in[gid].y * norm;

        out[gid] = result;
    }
}