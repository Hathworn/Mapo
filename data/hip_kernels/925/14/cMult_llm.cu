#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) 
                 + (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x 
                 + threadIdx.x;
    return threadId;
}

__global__ void cMult(double2* in1, double2* in2, double2* out) {
    unsigned int gid = getGid3d3d();
    
    // Load input data once into local registers for faster access
    double2 tin1 = in1[gid];
    double2 tin2 = in2[gid];

    // Calculate complex multiplication and store result
    out[gid].x = (tin1.x * tin2.x - tin1.y * tin2.y);
    out[gid].y = (tin1.x * tin2.y + tin1.y * tin2.x);
}