#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) 
                   + (threadIdx.y * blockDim.x) 
                   + (threadIdx.z * (blockDim.x * blockDim.y)) 
                   + threadIdx.x;
    return threadId;
}

__global__ void kring_rotation_Ax(double *x, double *y, double *z, double xMax, double yMax, double zMax, double omegaX, double omegaY, double omegaZ, double omega, double fudge, double *A){
    int gid = getGid3d3d();
    int xid = blockDim.x * blockIdx.x + threadIdx.x;
    int yid = blockDim.y * blockIdx.y + threadIdx.y;
    int zid = blockDim.z * blockIdx.z + threadIdx.z;

    // Pre-fetch data from global memory for reuse
    double xval = x[xid];
    double yval = y[yid];
    double zval = z[zid];

    // Calculate angle only once
    double theta = atan2(yval, xval);

    // Use computed values to update A
    A[gid] = (zval + zMax) * cos(theta) * omega * omegaX;
}