#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.z * blockDim.x * blockDim.y) // Optimize thread index calculation
                 + (threadIdx.y * blockDim.x)
                 + threadIdx.x;
    return threadId;
}

__global__ void thread_test(double *in, double *out){
    unsigned int Gid = getGid3d3d();

    // Write Gid to the output array
    out[Gid] = static_cast<double>(Gid); // Use explicit type cast for consistency
}