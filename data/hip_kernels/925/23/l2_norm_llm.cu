#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                 + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
    return threadId;
}

__global__ void l2_norm(double2 *in1, double2 *in2, double *out) {
    // Calculate global thread ID once
    int gid = getGid3d3d();
    double2 a = in1[gid];
    double2 b = in2[gid];
    // Compute L2 norm using shared temporary variables
    double sum = a.x * a.x + a.y * a.y + b.x * b.x + b.y * b.y;
    out[gid] = sqrt(sum);
}