#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.y * blockDim.x) 
                   + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__device__ double2 pow(double2 a, int b) {
    double r = sqrt(a.x * a.x + a.y * a.y);
    double theta = atan2(a.y, a.x); // atan2 for correct angle calculation
    return { pow(r, b) * cos(b * theta), pow(r, b) * sin(b * theta) };
}

__global__ void ktorus_wfc(double *x, double *y, double *z, double *items, double winding, double *phi, double2 *wfc) {
    int xid = blockIdx.x * blockDim.x + threadIdx.x;
    int yid = blockIdx.y * blockDim.y + threadIdx.y;
    int zid = blockIdx.z * blockDim.z + threadIdx.z;
    int gid = (zid * gridDim.y * gridDim.x + yid * gridDim.x + xid); // More intuitive gid calculation

    // Combine adjacent operations to reduce computation and increase clarity
    double dx = x[xid] - items[6];
    double dy = y[yid] - items[7];
    double rad = sqrt(dx * dx + dy * dy) - 0.5 * items[0];

    double rad_norm = pow(rad / (items[14] * items[15] * 0.5), 2);
    double z_norm = pow(z[zid] / (items[14] * items[17] * 0.5), 2);
    
    // Precompute normalization factor to save recalculation inside exp()
    double norm_factor = rad_norm + z_norm;

    wfc[gid].x = exp(-norm_factor);
    wfc[gid].y = 0.0;
}