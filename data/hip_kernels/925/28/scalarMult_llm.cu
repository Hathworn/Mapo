#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void scalarMult(double2* in, double factor, double2* out){
    // Calculate global thread ID directly, avoiding repeated operations
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x 
                     + (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x
                     + (blockIdx.z * blockDim.z + threadIdx.z) * gridDim.x * gridDim.y * blockDim.x * blockDim.y;
    
    // Perform multiplication and store result using fewer temporary variables
    out[gid].x = in[gid].x * factor;
    out[gid].y = in[gid].y * factor;
}