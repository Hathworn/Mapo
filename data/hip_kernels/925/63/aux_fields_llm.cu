#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.y * blockDim.x) + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void aux_fields(double *V, double *K, double gdt, double dt, double* Ax, double *Ay, double* Az, double *px, double *py, double *pz, double* pAx, double* pAy, double* pAz, double2* GV, double2* EV, double2* GK, double2* EK, double2* GpAx, double2* GpAy, double2* GpAz, double2* EpAx, double2* EpAy, double2* EpAz){
    int gid = getGid3d3d();
  
    // Pre-compute values that are reused
    double gdt_HBAR = gdt / HBAR;
    double dt_HBAR = dt / HBAR;
  
    // Calculate GV and GK using pre-computed values
    GV[gid].x = exp(-V[gid]*(gdt/(2*HBAR)));
    GK[gid].x = exp(-K[gid]*gdt_HBAR);
    GV[gid].y = 0.0;
    GK[gid].y = 0.0;

    // Pre-compute Ax, Ay, Az values
    double px_val = Ax[gid] * px[threadIdx.x];
    double py_val = Ay[gid] * py[threadIdx.y];
    double pz_val = Az[gid] * pz[threadIdx.z];
  
    pAx[gid] = px_val;
    pAy[gid] = py_val;
    pAz[gid] = pz_val;
  
    // Calculate GpAx, GpAy, GpAz using pre-computed values
    GpAx[gid].x = exp(-px_val * gdt);
    GpAx[gid].y = 0;
    GpAy[gid].x = exp(-py_val * gdt);
    GpAy[gid].y = 0;
    GpAz[gid].x = exp(-pz_val * gdt);
    GpAz[gid].y = 0;

    // Calculate EV, EK, EpAx, EpAy, EpAz using pre-computed values
    double common_factor = V[gid] * (dt / (2 * HBAR));
    EV[gid].x = cos(-common_factor);
    EV[gid].y = sin(-common_factor);
  
    common_factor = K[gid] * dt_HBAR;
    EK[gid].x = cos(-common_factor);
    EK[gid].y = sin(-common_factor);
  
    common_factor = pAz[gid] * dt;
    EpAz[gid].x = cos(-common_factor);
    EpAz[gid].y = sin(-common_factor);
  
    common_factor = pAy[gid] * dt;
    EpAy[gid].x = cos(-common_factor);
    EpAy[gid].y = sin(-common_factor);
  
    common_factor = pAx[gid] * dt;
    EpAx[gid].x = cos(-common_factor);
    EpAx[gid].y = sin(-common_factor);
}