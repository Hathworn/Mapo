#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "aux_fields_llm.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    double *V = NULL;
hipMalloc(&V, XSIZE*YSIZE);
double *K = NULL;
hipMalloc(&K, XSIZE*YSIZE);
double gdt = 1;
double dt = 1;
double *Ax = NULL;
hipMalloc(&Ax, XSIZE*YSIZE);
double *Ay = NULL;
hipMalloc(&Ay, XSIZE*YSIZE);
double *Az = NULL;
hipMalloc(&Az, XSIZE*YSIZE);
double *px = NULL;
hipMalloc(&px, XSIZE*YSIZE);
double *py = NULL;
hipMalloc(&py, XSIZE*YSIZE);
double *pz = NULL;
hipMalloc(&pz, XSIZE*YSIZE);
double *pAx = NULL;
hipMalloc(&pAx, XSIZE*YSIZE);
double *pAy = NULL;
hipMalloc(&pAy, XSIZE*YSIZE);
double *pAz = NULL;
hipMalloc(&pAz, XSIZE*YSIZE);
double2 *GV = NULL;
hipMalloc(&GV, XSIZE*YSIZE);
double2 *EV = NULL;
hipMalloc(&EV, XSIZE*YSIZE);
double2 *GK = NULL;
hipMalloc(&GK, XSIZE*YSIZE);
double2 *EK = NULL;
hipMalloc(&EK, XSIZE*YSIZE);
double2 *GpAx = NULL;
hipMalloc(&GpAx, XSIZE*YSIZE);
double2 *GpAy = NULL;
hipMalloc(&GpAy, XSIZE*YSIZE);
double2 *GpAz = NULL;
hipMalloc(&GpAz, XSIZE*YSIZE);
double2 *EpAx = NULL;
hipMalloc(&EpAx, XSIZE*YSIZE);
double2 *EpAy = NULL;
hipMalloc(&EpAy, XSIZE*YSIZE);
double2 *EpAz = NULL;
hipMalloc(&EpAz, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
aux_fields<<<gridBlock, threadBlock>>>(V,K,gdt,dt,Ax,Ay,Az,px,py,pz,pAx,pAy,pAz,GV,EV,GK,EK,GpAx,GpAy,GpAz,EpAx,EpAy,EpAz);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
aux_fields<<<gridBlock, threadBlock>>>(V,K,gdt,dt,Ax,Ay,Az,px,py,pz,pAx,pAy,pAz,GV,EV,GK,EK,GpAx,GpAy,GpAz,EpAx,EpAy,EpAz);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
aux_fields<<<gridBlock, threadBlock>>>(V,K,gdt,dt,Ax,Ay,Az,px,py,pz,pAx,pAy,pAz,GV,EV,GK,EK,GpAx,GpAy,GpAz,EpAx,EpAy,EpAz);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}