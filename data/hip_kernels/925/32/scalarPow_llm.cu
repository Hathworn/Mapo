#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) +
                   (threadIdx.y * blockDim.x) +
                   (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__device__ double2 pow(double2 a, int b) {
    double r = sqrt(a.x * a.x + a.y * a.y);
    double theta = atan2(a.y, a.x); // Use atan2 for correct quadrant
    return {pow(r, b) * cos(b * theta), pow(r, b) * sin(b * theta)};
}

__global__ void scalarPow(double2* in, double param, double2* out) {
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x; // Simplified linear index
    double2 val = in[gid];
    double r = sqrt(val.x * val.x + val.y * val.y);
    double theta = atan2(val.y, val.x); // Use atan2 for correct quadrant
    double mag = pow(r, param);
    out[gid].x = mag * cos(param * theta);
    out[gid].y = mag * sin(param * theta);
}