#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) + (threadIdx.y * blockDim.x) + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void vecMult(double *in, double *factor, double *out) {
    unsigned int gid = getGid3d3d();
    // Direct calculation and output without use of intermediate variable
    out[gid] = in[gid] * factor[gid];
}