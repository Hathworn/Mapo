#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) 
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) 
                 + threadIdx.x;
    return threadId;
}

__global__ void complexAbsSum(double2 *in1, double2 *in2, double *out) {
    int gid = getGid3d3d();
    
    // Utilize shared memory to reduce global memory access for better performance
    __shared__ double2 sharedMem1[256]; // Assuming a maximum block size of 256 threads
    __shared__ double2 sharedMem2[256];

    // Load data from global to shared memory
    sharedMem1[threadIdx.x] = in1[gid];
    sharedMem2[threadIdx.x] = in2[gid];
    __syncthreads();

    // Compute sum and magnitude in shared memory
    double2 temp;
    temp.x = sharedMem1[threadIdx.x].x + sharedMem2[threadIdx.x].x;
    temp.y = sharedMem1[threadIdx.x].y + sharedMem2[threadIdx.x].y;
    out[gid] = sqrt(temp.x * temp.x + temp.y * temp.y);
}