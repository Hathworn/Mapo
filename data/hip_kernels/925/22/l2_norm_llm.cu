#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                  + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                   + (threadIdx.y * blockDim.x)
                   + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void l2_norm(double *in1, double *in2, double *out) {
    // Use built-in function to improve performance of this kernel
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    double val1 = in1[gid];
    double val2 = in2[gid];
    out[gid] = sqrt(val1 * val1 + val2 * val2);
}