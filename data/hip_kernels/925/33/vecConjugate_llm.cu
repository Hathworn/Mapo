#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void vecConjugate(double2 *in, double2 *out) {
    unsigned int gid = getGid3d3d();
    if (gid < gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z) { // Ensure thread is within bounds
        double2 input = in[gid];
        out[gid] = make_double2(input.x, -input.y); // Use intrinsic function for double2 assignment
    }
}