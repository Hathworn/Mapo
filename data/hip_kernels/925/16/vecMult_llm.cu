#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                 + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void vecMult(double2 *in, double *factor, double2 *out){
    unsigned int gid = getGid3d3d();
    
    // Load input values to registers to avoid repeated global memory access
    double2 inputVal = in[gid];
    double factorVal = factor[gid];
    
    // Perform computation in registers
    double2 result;
    result.x = inputVal.x * factorVal;
    result.y = inputVal.y * factorVal;
    
    // Write result back to global memory
    out[gid] = result;
}