#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void is_eq(bool *a, bool *b, bool *ans){
    // Initialize thread-specific answer
    int gid = getGid3d3d();
    if (a[gid] != b[gid]){
        ans[gid] = false;  // Store result per thread
    } else {
        ans[gid] = true;
    }
}