#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void kstd_wfc(double *x, double *y, double *z, double *items, double winding, double *phi, double2 *wfc) {
    int gid = getGid3d3d();
    int xid = blockDim.x * blockIdx.x + threadIdx.x;
    int yid = blockDim.y * blockIdx.y + threadIdx.y;
    int zid = blockDim.z * blockIdx.z + threadIdx.z;

    // Precompute reused values
    double inv_item14_sq = 1.0 / (items[14] * items[14]);
    double inv_items15_sq = 1.0 / (items[15] * items[15]);
    double inv_items16_sq = 1.0 / (items[16] * items[16]);
    double inv_items17_sq = 1.0 / (items[17] * items[17]);

    phi[gid] = -fmod(winding * atan2(y[yid], x[xid]), 2 * PI);

    double exponent = exp(-(x[xid] * x[xid] * inv_item14_sq * inv_items15_sq
                          + y[yid] * y[yid] * inv_item14_sq * inv_items16_sq
                          + z[zid] * z[zid] * inv_item14_sq * inv_items17_sq));
    // Reduce redundant computations
    wfc[gid].x = exponent * cos(phi[gid]);
    wfc[gid].y = -exponent * sin(phi[gid]);
}