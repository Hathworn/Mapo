#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double complexMagnitude(double2 in) {
    return sqrt(in.x * in.x + in.y * in.y);
}

// Use faster calculation for global thread ID
__device__ unsigned int getGid3d3d() {
    return threadIdx.x + blockIdx.x * blockDim.x + 
           (threadIdx.y + blockIdx.y * blockDim.y) * gridDim.x * blockDim.x + 
           (threadIdx.z + blockIdx.z * blockDim.z) * gridDim.x * gridDim.y * blockDim.x * blockDim.y;
}

// Optimize kernel by avoiding redundant index calculation and using memory coalescing
__global__ void complexMagnitudeSquared(double2 *in, double *out) {
    unsigned int gid = getGid3d3d();
    double2 val = in[gid]; // Access input once
    out[gid] = val.x * val.x + val.y * val.y; // Perform squared magnitude operation
}