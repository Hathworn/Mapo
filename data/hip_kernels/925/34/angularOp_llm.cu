#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + threadIdx.x + threadIdx.y * blockDim.x
                 + threadIdx.z * (blockDim.x * blockDim.y);
    return threadId;
}

__global__ void angularOp(double omega, double dt, double2* wfc, double* xpyypx, double2* out){
    unsigned int gid = getGid3d3d();
    double op = exp(-omega * xpyypx[gid] * dt); // Compute once and reuse
    double2 wf = wfc[gid]; // Reduce global memory access
    double2 result;
    result.x = wf.x * op;
    result.y = wf.y * op;
    out[gid] = result;
}
```
