#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void vecSum(double *in, double *factor, double *out) {
    unsigned int gid = getGid3d3d();
    // Load inputs into registers to reduce global memory accesses
    double inVal = in[gid];
    double factorVal = factor[gid];
    // Perform the operation and store the result
    out[gid] = inVal + factorVal;
}