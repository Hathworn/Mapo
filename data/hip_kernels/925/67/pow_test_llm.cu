#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double2 pow_double2(double2 a, int b) {
    // Use more numerically stable calculations
    double r = hypot(a.x, a.y);
    double theta = atan2(a.y, a.x);
    double rb = pow(r, b);
    double btheta = b * theta;
    return {rb * cos(btheta), rb * sin(btheta)};
}

__global__ void pow_test(double2* a, int b, double2* c) {
    // Perform computation with improved pow function
    c[0] = pow_double2(a[0], b);
}