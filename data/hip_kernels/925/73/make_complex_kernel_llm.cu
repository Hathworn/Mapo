#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double2 make_complex(double in, int evolution_type) {
    double2 result;

    switch(evolution_type) {
        case 0: // No change
            result.x = in;
            result.y = 0;
            break;
        case 1: // Im. Time evolution
            result.x = exp(-in);
            result.y = 0;
            break;
        case 2: // Real Time evolution
            result.x = cos(-in);
            result.y = sin(-in);
            break;
    }

    return result;
}

__global__ void make_complex_kernel(double *in, int *evolution_type, double2 *out) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure thread does not exceed array bounds
    if (id < 3) {
        out[id] = make_complex(in[id], evolution_type[id]);
    }
}