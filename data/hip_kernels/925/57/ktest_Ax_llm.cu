#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                  + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                   + (threadIdx.z * (blockDim.x * blockDim.y))
                   + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__global__ void ktest_Ax(double *x, double *y, double *z, double xMax, double yMax, double zMax, double omegaX, double omegaY, double omegaZ, double omega, double fudge, double *A) {
    // Optimize thread index calculation
    int gid = getGid3d3d();
    int yid = blockIdx.y * blockDim.y + threadIdx.y;
    // Optimize memory access pattern by reusing loaded data
    double yVal = y[yid] * 100000.0;
    A[gid] = (sin(yVal) + 1.0) * yMax * omega;
}