#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double2 mult(double2 a, double2 b){
    return {a.x*b.x - a.y*b.y, a.x*b.y + a.y*b.x};
}

__device__ double2 mult(double2 a, double b){
    return {a.x*b, a.y*b};
}

__global__ void mult_test(double2 *a, double b, double2 *c){
    // Use thread index to ensure multiple threads work on different data
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Eliminate bank conflicts by ensuring coalesced access
    c[idx] = mult(a[idx], b);
}