#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getTid3d3d() {
    return threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
}

__device__ unsigned int getBid3d3d() {
    return blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
}

__device__ unsigned int getGid3d3d() {
    return (blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z) * (blockDim.x * blockDim.y * blockDim.z) + threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
}

__device__ double2 mult(double2 a, double2 b) {
    return {a.x * b.x - a.y * b.y, a.x * b.y + a.y * b.x};
}

__device__ double2 mult(double2 a, double b) {
    return {a.x * b, a.y * b};
}

__global__ void multipass(double* input, double* output) {
    unsigned int tid = getTid3d3d();
    unsigned int gid = getGid3d3d();
    extern __shared__ double sdatad[];

    sdatad[tid] = input[gid];
    __syncthreads();

    // Use warp-efficient reduction
    for (int i = blockDim.x * blockDim.y * blockDim.z >> 1; i > 32; i >>= 1) {
        if (tid < i) {
            sdatad[tid] += sdatad[tid + i];
        }
        __syncthreads();
    }

    // Final warp reduction
    if (tid < 32) {
        volatile double* vsmem = sdatad;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    if (tid == 0) {
        output[getBid3d3d()] = sdatad[0];
    }
}