#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double complexMagnitude(double2 in) {
    return sqrt(in.x * in.x + in.y * in.y);
}

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
                  + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                   + (threadIdx.y * blockDim.x)
                   + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void complexMagnitudeSquared(double2 *in, double2 *out) {
    // Calculate the global ID
    int gid = blockIdx.x * blockDim.x + threadIdx.x 
              + (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x 
              + (blockIdx.z * blockDim.z + threadIdx.z) * gridDim.x * blockDim.x * gridDim.y * blockDim.y;

    // Optimize by re-using computations
    double x = in[gid].x;
    double y = in[gid].y;
    out[gid].x = x * x + y * y; // Store magnitude squared
    out[gid].y = 0;             // Ensure imaginary part is zero
}