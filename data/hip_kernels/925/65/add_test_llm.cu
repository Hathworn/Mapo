#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ double2 add(double2 a, double2 b){
    return {a.x + b.x, a.y + b.y};
}

__global__ void add_test(double2 *a, double2 *b, double2 *c){
    // Calculate unique index for the thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform addition only if index is within bounds
    if (idx == 0) {
        c[idx] = add(a[idx], b[idx]);
    }
}