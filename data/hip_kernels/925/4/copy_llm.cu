#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) 
                + (threadIdx.z * (blockDim.x * blockDim.y))
                + (threadIdx.y * blockDim.x)
                + threadIdx.x;
    return threadId;
}

// Optimized kernel function
__global__ void copy(double2 *in, double2 *out) {
    int gid = getGid3d3d();
    // Use shared memory to optimize data access. Assume grid and block sizes are manageable.
    __shared__ double2 sharedMem[1024]; // Adjust size according to block size constraints
    sharedMem[threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y] = in[gid];
    __syncthreads(); // Synchronize threads within block
    out[gid] = sharedMem[threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y];
}