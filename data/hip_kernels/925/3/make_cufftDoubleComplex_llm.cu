#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
    + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
    + (threadIdx.y * blockDim.x)
    + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}
__global__ void make_cufftDoubleComplex(double *in, double2 *out){
    int gid = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D grid to access gid
    int planeSize = blockDim.x * gridDim.x; // Calculate plane size
    int sliceOffset = blockIdx.y * gridDim.x * blockDim.x; // Calculate offset for slice
    int totalOffset = sliceOffset + gid; // Calculate total offset
    out[totalOffset].x = in[totalOffset]; // Map input to output using total offset
    out[totalOffset].y = 0;
}