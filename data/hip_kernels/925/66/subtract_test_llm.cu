#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double2 subtract(double2 a, double2 b) {
    return {a.x - b.x, a.y - b.y};
}

__global__ void subtract_test(double2 *a, double2 *b, double2 *c) {
    // Optimize by utilizing thread index for parallel computation
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    c[index] = subtract(a[index], b[index]);
}