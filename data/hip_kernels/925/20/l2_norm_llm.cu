#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) +
                   (threadIdx.y * blockDim.x) +
                   (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void l2_norm(double *in1, double *in2, double *in3, double *out) {
    // Use shared memory to store inputs
    extern __shared__ double sharedMem[];

    int local_tid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * (blockDim.x * blockDim.y);
    int global_tid = getGid3d3d();

    // Load inputs into shared memory for better memory access pattern
    sharedMem[local_tid] = in1[global_tid];
    sharedMem[local_tid + blockDim.x * blockDim.y * blockDim.z] = in2[global_tid];
    sharedMem[local_tid + 2 * blockDim.x * blockDim.y * blockDim.z] = in3[global_tid];
    __syncthreads();

    // Compute l2 norm using shared memory
    double val1 = sharedMem[local_tid];
    double val2 = sharedMem[local_tid + blockDim.x * blockDim.y * blockDim.z];
    double val3 = sharedMem[local_tid + 2 * blockDim.x * blockDim.y * blockDim.z];
    out[global_tid] = sqrt(val1 * val1 + val2 * val2 + val3 * val3);
}