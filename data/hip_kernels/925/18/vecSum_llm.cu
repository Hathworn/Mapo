#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.z * (blockDim.x * blockDim.y))
                 + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

__global__ void vecSum(double2 *in, double *factor, double2 *out) {
    unsigned int gid = getGid3d3d();

    // Cache the factor to reduce global memory access
    double f = factor[gid];
    
    // Perform computation with cached value
    double2 input = in[gid];
    double2 result;
    result.x = input.x + f;
    result.y = input.y + f;
    
    out[gid] = result;
}