#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
        + threadIdx.z * (blockDim.x * blockDim.y) 
        + threadIdx.y * blockDim.x + threadIdx.x;
    return threadId;
}

__global__ void kring_rotation_Az(
    double *x, double *y, double *z, 
    double xMax, double yMax, double zMax, 
    double omegaX, double omegaY, double omegaZ, 
    double omega, double fudge, double *A) 
{
    int xid = blockDim.x * blockIdx.x + threadIdx.x;
    int yid = blockDim.y * blockIdx.y + threadIdx.y;
    if (xid < xMax && yid < yMax) { // Boundary check to avoid out-of-bounds access
        double r = sqrt(x[xid] * x[xid] + y[yid] * y[yid]);
        int gid = getGid3d3d(); 
        A[gid] = r * omega * omegaX;
    }
}