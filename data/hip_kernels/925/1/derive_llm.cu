#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void derive(double2 *data, double2 *out, int stride, int gsize, double dx) {
    int gid = getGid3d3d();
    if (gid < gsize) {
        // Using shared memory for better memory access patterns
        __shared__ double2 sharedData[1024];  // Assumption: adjust the size based on blockDim
        sharedData[threadIdx.x] = data[gid];
        
        __syncthreads();  // Synchronize to ensure data is available
        
        double2 curVal = sharedData[threadIdx.x];

        if (gid + stride < gsize) {
            double2 nextVal = (threadIdx.x + stride < 1024) ? sharedData[threadIdx.x + stride] : data[gid + stride];
            out[gid].x = (nextVal.x - curVal.x) / dx;
            out[gid].y = (nextVal.y - curVal.y) / dx;
        } else {
            out[gid].x = curVal.x / dx;
            out[gid].y = curVal.y / dx;
        }
    }
}