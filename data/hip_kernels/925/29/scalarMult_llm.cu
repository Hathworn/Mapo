#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d(){
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                  + (threadIdx.y * blockDim.x)
                  + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void scalarMult(double* __restrict__ in, double factor, double* __restrict__ out){
    unsigned int gid = getGid3d3d();
    // Use registers for intermediate computation
    double result = in[gid] * factor;
    // Minimize memory writes
    out[gid] = result;
}