#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.y * blockDim.x)
                 + (threadIdx.z * (blockDim.x * blockDim.y)) + threadIdx.x;
    return threadId;
}

__global__ void kharmonic_V(double *x, double *y, double *z, double* items, double *Ax, double *Ay, double *Az, double *V) {

    int gid = getGid3d3d();
    int xid = blockDim.x * blockIdx.x + threadIdx.x;
    int yid = blockDim.y * blockIdx.y + threadIdx.y;
    int zid = blockDim.z * blockIdx.z + threadIdx.z;

    // Load items variables into registers to reduce global memory access
    double item3 = items[3];
    double item4 = items[4];
    double item5 = items[5];
    double item6 = items[6];
    double item7 = items[7];
    double item8 = items[8];
    double item9 = items[9];
    double item10 = items[10];
    double item11 = items[11];

    // Calculate potential energy components
    double V_x = item3 * (x[xid] + item6);
    double V_y = item10 * item4 * (y[yid] + item7);
    double V_z = item11 * item5 * (z[zid] + item8);

    // Compute the total potential energy with reduced operations
    double potential_energy = V_x * V_x + V_y * V_y + V_z * V_z;
    double magnetic_energy = Ax[gid] * Ax[gid] + Ay[gid] * Ay[gid] + Az[gid] * Az[gid];

    V[gid] = 0.5 * item9 * (potential_energy + magnetic_energy);
}