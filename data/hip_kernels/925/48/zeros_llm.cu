#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void zeros(double *field, int n) {
    int xid = blockDim.x * blockIdx.x + threadIdx.x;
    // Use coalesced memory access by ensuring contiguous threads 
    // access contiguous memory locations
    for (int i = xid; i < n; i += blockDim.x * gridDim.x) {
        field[i] = 0;
    }
}