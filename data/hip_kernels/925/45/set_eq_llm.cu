#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
                 + (threadIdx.z * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
    return threadId;
}

__global__ void set_eq(double *in1, double *in2) {
    int gid = getGid3d3d();
    if (gid < gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z)  // Boundary check
        in2[gid] = in1[gid];
}