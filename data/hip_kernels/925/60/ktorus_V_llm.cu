#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int getGid3d3d() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z) +
                   (threadIdx.y * blockDim.x) +
                   (threadIdx.z * (blockDim.x * blockDim.y)) + 
                   threadIdx.x;
    return threadId;
}

__global__ void ktorus_V(double *x, double *y, double *z, double* items, double *Ax, double *Ay, double *Az, double *V) {
    int gid = getGid3d3d();
    int xid = blockDim.x * blockIdx.x + threadIdx.x;
    int yid = blockDim.y * blockIdx.y + threadIdx.y;
    int zid = blockDim.z * blockIdx.z + threadIdx.z;

    double deltaX = x[xid] - items[6];
    double deltaY = y[yid] - items[7];
    double rad = sqrt(deltaX * deltaX + deltaY * deltaY) - 0.5 * items[0];
    
    double omegaR = items[3] * items[3] + items[4] * items[4];
    double deltaZ = z[zid] - items[8];
    double V_tot = 2 * items[5] * items[5] * deltaZ * deltaZ + 
                   omegaR * (rad * rad + items[12] * rad * z[zid]);
    double AxVal = Ax[gid], AyVal = Ay[gid], AzVal = Az[gid];
    
    // Optimized calculation reduced repetitive memory accesses
    V[gid] = 0.5 * items[9] * (V_tot + 
                               AxVal * AxVal + 
                               AyVal * AyVal + 
                               AzVal * AzVal);
}