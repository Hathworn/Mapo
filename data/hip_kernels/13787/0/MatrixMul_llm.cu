#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
/*CUDA 2-D Matrix Multiplication*/

#define TILE_WIDTH 16 // Optimize TILE_WIDTH for better performance
#define WIDTH 100

// main routine
__global__ void MatrixMul(float *A_d, float *B_d, float *C_d) {
    // Calculate global thread id for the 2D grid
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Initialize variable to accumulate the sum for C_d[row, col]
    float sum = 0.0f;

    // Perform the dot product for this element
    for (int k = 0; k < WIDTH; k++) {
        sum += A_d[row * WIDTH + k] * B_d[k * WIDTH + col];
    }

    // Write the result to the output matrix
    C_d[row * WIDTH + col] = sum;
}