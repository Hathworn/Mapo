#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    __shared__ int uni;

    if (threadIdx.x == 0) {
        uni = uniforms[blockIdx.x + blockOffset];
    }

    __syncthreads();

    unsigned int address = __mul24(blockIdx.x, (blockDim.x << 1)) + baseIndex + threadIdx.x;

    // First add, all threads
    g_data[address] += uni;

    // Second add, only threads within the valid range
    if ((threadIdx.x + blockDim.x) < n) {
        g_data[address + blockDim.x] += uni;
    }
}