#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = idx; i < N; i += stride) {
        a[i] = a[i] * a[i]; // Square elements using grid-stride loop
    }
}