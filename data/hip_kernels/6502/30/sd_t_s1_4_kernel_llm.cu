#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_4_kernel(int h1d, int h2d, int h3d, int p4d, int p5d, int p6d, int p4ld_t2, int h1ld_t2, int h3ld_v2, int h2ld_v2, int p6ld_v2, int p5ld_v2, int h3ld_t3, int h2ld_t3, int h1ld_t3, int p6ld_t3, int p5ld_t3, int p4ld_t3, double *t3d, double *t2_d, double *v2_d, int p4, int total_x) {
    int h1, h2, h3, p6, p5;
    __shared__ double t2_shm[T1 * 2 * Tcomm];

    for (int i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        if (i < h1d * p4d)
            t2_shm[i] = t2_d[i];
    }
    __syncthreads(); // Ensure shared memory is ready before proceeding

    int rest_x = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Calculate offset only once and use stride for iteration
    const int stride = gridDim.x * blockDim.x;
    for (int thread_x = rest_x; thread_x < total_x; thread_x += stride) {
        int temp = thread_x;
        
        h3 = temp % h3d;
        temp /= h3d;
        h2 = temp % h2d;
        temp /= h2d;
        p6 = temp % p6d;
        temp /= p6d;
        p5 = temp % p5d;

        for (h1 = 0; h1 < h1d; h1++) {
            for (p4 = 0; p4 < p4d; p4++) {
                // Avoid repetitive memory access by computing offsets once
                int t2_shm_index = h1 * p4d + p4;
                int v2_d_index = h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2 + p5 * p5ld_v2;
                int t3d_index = h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p5 * p5ld_t3 + p4 * p4ld_t3;

                t3d[t3d_index] -= t2_shm[t2_shm_index] * v2_d[v2_d_index];
            }
        }
    }
    __syncthreads();
}