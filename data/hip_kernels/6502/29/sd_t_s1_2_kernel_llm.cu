#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_2_kernel(int h1d, int h2d, int h3d, int p4d, int p6d, int p4ld_t2, int h1ld_t2, int h3ld_v2, int h2ld_v2, int p6ld_v2, int h3ld_t3, int h2ld_t3, int h1ld_t3, int p6ld_t3, int p4ld_t3, double *t2_d, double *v2_d, int p4, int total_x, double* t3d) {
    int h1, h2, h3, p6;
    __shared__ double t2_shm[T1 * 2 * Tcomm];
    
    // Load t2_d into shared memory in a coalesced manner
    int t2_shm_index = threadIdx.x;
    while (t2_shm_index < h1d * p4d) {
        t2_shm[t2_shm_index] = t2_d[t2_shm_index];
        t2_shm_index += blockDim.x;
    }
    __syncthreads();
    
    // Calculate global thread index
    int rest_x = blockIdx.x;
    int thread_x = T2 * T1 * rest_x + threadIdx.x;
    
    // Loop over the grid to ensure all elements are processed
    for (int i = 0; i < total_x; i += gridDim.x * blockDim.x) {
        if (thread_x + i >= total_x) continue;  // Skip out-of-bound threads

        // Calculate indices h3, h2, h1, p6
        int idx = thread_x + i;
        h3 = idx % h3d;
        idx /= h3d;
        h2 = idx % h2d;
        idx /= h2d;
        p6 = idx % p6d;

        // Iterate over h1 and p4 and update t3d
        for (h1 = 0; h1 < h1d; h1++) {
            for (p4 = 0; p4 < p4d; p4++) {
                atomicSub(&t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3],
                          t2_shm[h1 * p4d + p4] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2]);
            }
        }
    }
    __syncthreads();
}