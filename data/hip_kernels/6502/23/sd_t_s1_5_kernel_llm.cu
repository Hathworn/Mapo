#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sd_t_s1_5_kernel(size_t h1d, size_t h2d, size_t h3d, size_t p4d, size_t p5d, size_t p6d, size_t p4ld_t2, size_t h1ld_t2, size_t h3ld_v2, size_t h2ld_v2, size_t p6ld_v2, size_t p5ld_v2, size_t h3ld_t3, size_t h2ld_t3, size_t h1ld_t3, size_t p6ld_t3, size_t p5ld_t3, size_t p4ld_t3, double *t3d, double *t2_d, double *v2_d, size_t p4, size_t total_x) {
    size_t h1, h2, h3, p6, p5;
    __shared__ double t2_shm[T1*4*Tcomm];

    // Optimize shared memory loading with a single loop
    for (size_t i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        t2_shm[i] = t2_d[i];
    }
    __syncthreads();

    size_t thread_x = blockIdx.x * blockDim.x + threadIdx.x;

    // Improve loop order and bounds for better memory access
    for (size_t idx = thread_x; idx < total_x; idx += gridDim.x * blockDim.x) {
        size_t rest_x = idx;
        h3 = rest_x % h3d; rest_x /= h3d;
        h2 = rest_x % h2d; rest_x /= h2d;
        p6 = rest_x % p6d; rest_x /= p6d;
        p5 = rest_x % p5d;
        
        for (h1 = 0; h1 < h1d; h1++) {
            for (p4 = 0; p4 < p4d; p4++) {
                t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p5 * p5ld_t3 + p4 * p4ld_t3] += 
                    t2_shm[h1 * p4d + p4] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2 + p5 * p5ld_v2];
            }
        }
    }
    __syncthreads();
}