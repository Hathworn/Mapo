#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define T1 128  // Example value for T1
#define T2 128  // Example value for T2
#define Tcomm 128  // Example value for Tcomm

__global__ void sd_t_s1_4_kernel(size_t h1d, size_t h2d, size_t h3d, size_t p4d, size_t p5d, size_t p6d, size_t p4ld_t2, size_t h1ld_t2, size_t h3ld_v2, size_t h2ld_v2, size_t p6ld_v2, size_t p5ld_v2, size_t h3ld_t3, size_t h2ld_t3, size_t h1ld_t3, size_t p6ld_t3, size_t p5ld_t3, size_t p4ld_t3, double *t3d, double *t2_d, double *v2_d, size_t p4, size_t total_x) {
    size_t h1,h2,h3,p6,p5;
    __shared__ double t2_shm[T1*4*Tcomm];

    // Load t2_d data into shared memory
    for (size_t i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        if (i < h1d * p4d) {
            t2_shm[i] = t2_d[i];
        }
    }
    
    size_t rest_x = blockIdx.x;
    size_t thread_x = T2 * T1 * rest_x + threadIdx.x;
    rest_x = thread_x;
    __syncthreads();

    // Main loop for computation
    for (size_t i = thread_x; i < total_x; i += gridDim.x * blockDim.x) {
        rest_x = i;
        h3 = rest_x % h3d;
        rest_x /= h3d;
        h2 = rest_x % h2d;
        rest_x /= h2d;
        p6 = rest_x % p6d;
        rest_x /= p6d;
        p5 = rest_x % p5d;

        if (i < total_x) {
            for (h1 = 0; h1 < h1d; h1++) {
                for (p4 = 0; p4 < p4d; p4++) {
                    // Use indices directly for computation reduction
                    double t2_value = t2_shm[h1 * p4d + p4];
                    double v2_value = v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2 + p5 * p5ld_v2];
                    t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p5 * p5ld_t3 + p4 * p4ld_t3] -= t2_value * v2_value;
                }
            }
        }
    }
    __syncthreads();
}