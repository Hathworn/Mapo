#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 256

__global__ void sd_t_s1_1_kernel(size_t h1d, size_t h2d, size_t h3d, size_t p4d, size_t p6d,
                                 size_t p4ld_t2, size_t h1ld_t2, size_t h3ld_v2, size_t h2ld_v2,
                                 size_t p6ld_v2, size_t h3ld_t3, size_t h2ld_t3, size_t h1ld_t3,
                                 size_t p6ld_t3, size_t p4ld_t3, double *t2_d, double *v2_d,
                                 size_t p4, size_t total_x, double* t3d) {
    size_t h1, h2, h3, p6;
    __shared__ double t2_shm[BLOCK_SIZE];

    for (size_t i = threadIdx.x; i < h1d * p4d; i += BLOCK_SIZE) {
        if (i < h1d * p4d) {
            t2_shm[i] = t2_d[i];  // Load shared memory only if within bounds
        }
    }
    __syncthreads();

    size_t global_id = blockIdx.x * blockDim.x + threadIdx.x;

    for (size_t i = global_id; i < total_x; i += gridDim.x * BLOCK_SIZE) {
        size_t rest_x = i;
        h3 = rest_x % h3d;
        rest_x /= h3d;
        h2 = rest_x % h2d;
        rest_x /= h2d;
        p6 = rest_x % p6d;

        for (h1 = 0; h1 < h1d; h1++) {
            for (p4 = 0; p4 < p4d; p4++) {
                atomicAdd(&t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3],
                          t2_shm[h1 * p4d + p4] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2]);
            }
        }
    }
    __syncthreads();
}