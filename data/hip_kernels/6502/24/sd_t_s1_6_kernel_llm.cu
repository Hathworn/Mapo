#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_6_kernel(size_t h1d, size_t h2d, size_t h3d, size_t p4d, size_t p5d, size_t p6d, size_t p4ld_t2, size_t h1ld_t2, size_t h3ld_v2, size_t h2ld_v2, size_t p6ld_v2, size_t p5ld_v2, size_t h3ld_t3, size_t h2ld_t3, size_t h1ld_t3, size_t p6ld_t3, size_t p5ld_t3, size_t p4ld_t3, double *t3d, double *t2_d, double *v2_d, size_t p4, size_t total_x) {
    size_t h1, h2, h3, p6, p5;
    __shared__ double t2_shm[T1 * 4 * Tcomm];

    // Load t2_d into shared memory efficiently
    for(size_t i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        if(i < h1d * p4d)
            t2_shm[i] = t2_d[i];
    }

    size_t rest_x = blockIdx.x * blockDim.x + threadIdx.x;
    __syncthreads();  // Ensure all data is loaded before computation

    // Adjust loop to avoid exceeding total_x
    while (rest_x < total_x) {
        size_t tmp = rest_x;
        h3 = tmp % h3d; tmp /= h3d;
        h2 = tmp % h2d; tmp /= h2d;
        p6 = tmp % p6d; tmp /= p6d;
        p5 = tmp % p5d;

        for(h1 = 0; h1 < h1d; h1++) {
            for(size_t p4_idx = 0; p4_idx < p4d; p4_idx++) {
                // Use loop unrolling or other optimizations here if beneficial
                t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p5 * p5ld_t3 + p4_idx * p4ld_t3] -=
                    t2_shm[h1 * p4d + p4_idx] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2 + p5 * p5ld_v2];
            }
        }
        rest_x += gridDim.x * blockDim.x;  // Move to the next batch
    }
    __syncthreads();
}