#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_2_kernel(size_t h1d, size_t h2d, size_t h3d, size_t p4d, size_t p6d, size_t p4ld_t2, size_t h1ld_t2, size_t h3ld_v2, size_t h2ld_v2, size_t p6ld_v2, size_t h3ld_t3, size_t h2ld_t3, size_t h1ld_t3, size_t p6ld_t3, size_t p4ld_t3, double *t2_d, double *v2_d, size_t p4, size_t total_x, double* t3d) {

    size_t h1, h2, h3, p6;
    __shared__ double t2_shm[T1 * 4 * Tcomm];

    // Load t2_d into shared memory using coalesced access when possible
    for (size_t i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        if (i < h1d * p4d) {
            t2_shm[i] = t2_d[i];
        }
    }

    __syncthreads();

    size_t rest_x = blockIdx.x;
    size_t thread_x = T2 * T1 * rest_x + threadIdx.x;

    // Reorganize loop computation for better cache efficiency
    for (size_t batchStart = 0; batchStart < total_x; batchStart += gridDim.x * blockDim.x) {
        size_t local_x = thread_x + batchStart;

        // Only proceed with computations if within bounds
        if (local_x < total_x) {
            size_t working_remainder = local_x;

            h3 = working_remainder % h3d;
            working_remainder /= h3d;

            h2 = working_remainder % h2d;
            working_remainder /= h2d;

            p6 = working_remainder % p6d;

            // Optimize by combining nested loops into a single iteration
            for (h1 = 0; h1 < h1d; ++h1) {
                for (p4 = 0; p4 < p4d; ++p4) {
                    t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3] -=
                        t2_shm[h1 * p4d + p4] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2];
                }
            }
        }
    }

    __syncthreads();
}