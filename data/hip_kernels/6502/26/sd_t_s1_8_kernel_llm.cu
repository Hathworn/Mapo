#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_8_kernel(size_t h1d, size_t h2d, size_t h3d, size_t p4d, size_t p6d, size_t p4ld_t2, size_t h1ld_t2, size_t h3ld_v2, size_t h2ld_v2, size_t p6ld_v2, size_t h3ld_t3, size_t h2ld_t3, size_t h1ld_t3, size_t p6ld_t3, size_t p4ld_t3, double *t3d, double *t2_d, double *v2_d, size_t p4, size_t total_x) {
    size_t h1, h2, h3, p6;
    __shared__ double t2_shm[T1 * 4 * Tcomm];
    
    // Load shared memory
    for(size_t i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        if(i < h1d * p4d)
            t2_shm[i] = t2_d[i];
    }
    
    size_t rest_x = blockIdx.x;
    size_t thread_x = T2 * T1 * rest_x + threadIdx.x;
    __syncthreads();
    
    // Loop unrolling for performance
    for(size_t i = 0; i < total_x; i += gridDim.x * blockDim.x) {
        size_t iter_x = thread_x + i;
        if(iter_x < total_x) {
            size_t temp_x = iter_x;
            h3 = temp_x % h3d;
            temp_x /= h3d;
            h2 = temp_x % h2d;
            temp_x /= h2d;
            p6 = temp_x % p6d;
            
            #pragma unroll // Unroll loops to improve performance
            for(h1 = 0; h1 < h1d; ++h1) {
                #pragma unroll
                for(p4 = 0; p4 < p4d; ++p4) {
                    // Perform the computation
                    t3d[h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3] -=
                        t2_shm[h1 * p4d + p4] * v2_d[h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2];
                }
            }
        }
    }
    __syncthreads();
}