#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sd_t_s1_1_kernel(int h1d, int h2d, int h3d, int p4d, int p6d, int p4ld_t2, int h1ld_t2, int h3ld_v2, int h2ld_v2, int p6ld_v2, int h3ld_t3, int h2ld_t3, int h1ld_t3, int p6ld_t3, int p4ld_t3, double *t2_d, double *v2_d, int p4, int total_x, double* t3d) {
    int h1, h2, h3, p6;
    __shared__ double t2_shm[T1 * 2 * Tcomm];

    // Load t2_d into shared memory
    for (int i = threadIdx.x; i < h1d * p4d; i += blockDim.x) {
        t2_shm[i] = t2_d[i];
    }
    __syncthreads(); // Ensure all t2_d is loaded before proceeding
    
    int thread_x = T2 * T1 * blockIdx.x + threadIdx.x;

    // Iterate over batches of work
    for (int i = thread_x; i < total_x; i += gridDim.x * blockDim.x) {
        int rest_x = i;
        h3 = rest_x % h3d;
        rest_x /= h3d;
        h2 = rest_x % h2d;
        rest_x /= h2d;
        p6 = rest_x % p6d;

        // Perform computation for this batch
        for (h1 = 0; h1 < h1d; h1++) {
            for (p4 = 0; p4 < p4d; p4++) {
                int t2_idx = h1 * p4d + p4;
                int v2_idx = h3 * h3ld_v2 + h2 * h2ld_v2 + p6 * p6ld_v2;
                int t3_idx = h3 * h3ld_t3 + h2 * h2ld_t3 + h1 * h1ld_t3 + p6 * p6ld_t3 + p4 * p4ld_t3;
                t3d[t3_idx] += t2_shm[t2_idx] * v2_d[v2_idx];
            }
        }
    }
    __syncthreads(); // Ensure all writes are done before kernel ends
}