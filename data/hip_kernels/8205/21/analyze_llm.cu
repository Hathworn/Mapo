#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void analyze(const float *input, float *sum, int numElements) {
    // Calculate global thread index for the flattened grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop through all elements handled by this thread
    for (int idx = i; idx < numElements; idx += blockDim.x * gridDim.x) {
        atomicAdd(sum + idx, input[idx]);
    }
}