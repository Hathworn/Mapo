#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum(const float *input, float *output, int numElements) {
    // Use shared memory to improve memory access speed
    extern __shared__ float sharedMem[];
    int tid = threadIdx.x;
    int index = tid + blockIdx.x * blockDim.x;
    float val = 0.0f;

    // Accumulate partial sums in each block
    for (int i = index; i < numElements; i += blockDim.x * gridDim.x) {
        val += input[i];
    }
    sharedMem[tid] = val;
    __syncthreads();

    // Reduce within the block
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedMem[tid] += sharedMem[tid + stride];
        }
        __syncthreads();
    }

    // Write block result to output
    if (tid == 0) {
        atomicAdd(output, sharedMem[0]);
    }
}