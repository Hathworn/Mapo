#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void init(uint32_t *v, uint32_t val, uint32_t n) {
    // Calculate the global thread index
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    // Assign 'val' to v[i] only if i is within bounds
    if (i < n) {
        v[i] = val;
    }

    // Use thread 0 of block 0 to print "init" once
    if (i == 0 && threadIdx.x == 0) {
        printf("init\n");
    }
}