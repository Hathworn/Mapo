#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int a, int b, int *c)
{
    // Use shared memory to optimize data access
    extern __shared__ int shared_data[];
    shared_data[0] = a;
    shared_data[1] = b;
    __syncthreads();
    
    // Perform addition using shared memory data
    if (threadIdx.x == 0) {
        *c = shared_data[0] + shared_data[1];
    }
}