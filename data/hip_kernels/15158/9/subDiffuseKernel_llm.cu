#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subDiffuseKernel(float *data, int x, int y, float pressure)
{
    // Calculate index only once for efficiency
    int index = NX * x + y;

    // Perform subtraction operation
    data[index] -= pressure;
}