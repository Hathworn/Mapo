#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void testKernel4(float *data1, float *data2)
{
    float t = 0.0f;
    float c = 0.0f;
    int index = NX * blockIdx.x + threadIdx.x;
    
    // Use shared memory for efficient data access
    extern __shared__ float sharedData[];
    sharedData[threadIdx.x] = data1[index];
    __syncthreads();

    if (threadIdx.x > 0)
    {
        t += sharedData[threadIdx.x - 1] - sharedData[threadIdx.x];
        c += 1.0f;
    }
    if (threadIdx.x < blockDim.x - 1)
    {
        t += sharedData[threadIdx.x + 1] - sharedData[threadIdx.x];
        c += 1.0f;
    }
    if (blockIdx.x > 0)
    {
        int prevBlockIndex = NX * (blockIdx.x - 1) + threadIdx.x;
        t += data1[prevBlockIndex] - sharedData[threadIdx.x];
        c += 1.0f;
    }
    if (blockIdx.x < gridDim.x - 1)
    {
        int nextBlockIndex = NX * (blockIdx.x + 1) + threadIdx.x;
        t += data1[nextBlockIndex] - sharedData[threadIdx.x];
        c += 1.0f;
    }

    if (blockIdx.x == 0)
        data2[index] = 1.0f;
    else
        data2[index] = data1[index] + t / c * DIFF_RATE;
}