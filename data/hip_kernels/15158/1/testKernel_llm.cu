#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel launches with many threads can clutter output and cause performance issues;
// Use a single thread to demonstrate functionality.
__global__ void testKernel()
{
    if (threadIdx.x == 0 && blockIdx.x == 0) { // Only one thread in one block prints
        printf("hi!\n");
    }
}