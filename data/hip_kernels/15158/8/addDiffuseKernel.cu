#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addDiffuseKernel(float *data, int x, int y, float pressure)
{
data[NX * x + y] += pressure;
}