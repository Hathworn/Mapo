#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addDiffuseKernel(float *data, int x, int y, float pressure)
{
    // Calculate global index to eliminate redundant memory access
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index == 0) {
        // Use pre-calculated index to update data
        data[NX * x + y] += pressure;
    }
}