#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void testKernel4r(float *data1, float *data2)
{
    float t = 0.0f;
    float c = 0.0f;
    int idx = NX * blockIdx.x + threadIdx.x; // Precompute index
    int idx_prev = NX * (blockIdx.x - 1) + threadIdx.x;
    int idx_next = NX * (blockIdx.x + 1) + threadIdx.x;

    if (blockIdx.x > 0) {
        t += (data2[idx_prev] - data2[idx]); // Use precomputed indices
        c += 1.0f;
    }
    if (blockIdx.x < NX - 1) {
        t += (data2[idx_next] - data2[idx]);
        c += 1.0f;
    }
    if (threadIdx.x > 0) {
        t += (data2[idx - 1] - data2[idx]);
        c += 1.0f;
    }
    if (threadIdx.x < NX - 1) {
        t += (data2[idx + 1] - data2[idx]);
        c += 1.0f;
    }

    if (blockIdx.x == 0)
        data1[idx] = 1.0f;
    else
        data1[idx] = data2[idx] + t / c * DIFF_RATE;

    return;
}