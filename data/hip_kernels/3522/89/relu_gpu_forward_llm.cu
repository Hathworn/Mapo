#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        // Use ternary operator for branch elimination
        out[tid] = max(0.0f, in[tid]);
    }
}