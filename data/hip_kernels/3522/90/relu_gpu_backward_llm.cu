#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    // Utilizing shared memory to reduce global memory traffic
    extern __shared__ float shared_indata[];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    // Load data into shared memory
    if (tid < N) {
        shared_indata[local_tid] = indata[tid];
    }
    __syncthreads();

    // Perform computation using shared memory
    if (tid < N) {
        ingrad[tid] = shared_indata[local_tid] > 0 ? outgrad[tid] : 0;
    }
}