#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread index '0' to avoid potential race condition
    if (threadIdx.x == 0) {
        *accuracy /= N;
    }
}