#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;

    // Use shared memory for efficient data access
    __shared__ float2 shared_data[CU1DBLOCK];
    int idx = thread_id;
    while (idx < half_length) {
        // Ignore special case
        if (idx != 0) {
            shared_data[thread_id] = reinterpret_cast<float2 *>(Ar)[idx];
            __syncthreads();

            float2 val = shared_data[thread_id];
            float ret = val.x * val.x + val.y * val.y;
            Aw[idx] = use_power ? ret : sqrtf(ret);
            __syncthreads();
        }
        idx += CU1DBLOCK;
    }

    // Handle special case with atomic operation for thread safety
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];
        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
    }
}