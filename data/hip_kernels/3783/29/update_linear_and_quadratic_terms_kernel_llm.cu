#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void update_linear_and_quadratic_terms_kernel(int32_t n, float prior_offset, float* cur_tot_weight, int32_t max_count, float* quadratic, float* linear) {
    float cur_weight = *cur_tot_weight;
    float val = 1.0f;

    if (max_count > 0.0f) {
        // Calculate new scale based on max_count and current weight
        float new_scale = max(cur_weight, static_cast<float>(max_count)) / max_count;
        // Adjust value based on scale change
        val += (new_scale - 1.0f);
    }

    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t stride = blockDim.x * gridDim.x;

    // Optimize loop by reducing redundant index calculation
    for (; idx < n; idx += stride) {
        int32_t diag_idx = ((idx + 1) * (idx + 2) / 2) - 1;
        quadratic[diag_idx] += val;
    }

    // Ensure only one thread updates linear[0]
    if (threadIdx.x == 0) {
        atomicAdd(&linear[0], val * prior_offset);
    }
}