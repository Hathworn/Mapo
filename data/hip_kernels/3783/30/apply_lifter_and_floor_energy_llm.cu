#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;

    float *feats = features + frame * ldf;

    // Apply lifter coefficients with shared memory to reduce global memory access
    if (cepstral_lifter != 0.0f) {
        extern __shared__ float shared_lifter_coeffs[];

        if (thread_id < num_cols && thread_id < blockDim.x) {
            shared_lifter_coeffs[thread_id] = lifter_coeffs[thread_id];
        }
        __syncthreads();

        for (int c = thread_id; c < num_cols; c += blockDim.x) {
            float lift = shared_lifter_coeffs[c];
            float f = feats[c];
            feats[c] = f * lift;
        }
    }

    // Thread 0 for each frame will apply energy optimization
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = log(energy_floor);

        energy = (energy_floor > 0.0f && energy < log_energy_floor) ? log_energy_floor : energy;
        feats[0] = energy;
    }
}