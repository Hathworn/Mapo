#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function for better memory access and execution efficiency
__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* __restrict__ feats, int32_t ldf, float* __restrict__ feats_sq, int32_t lds) {
    // Calculate row and column indices using block and thread indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundaries to ensure no out-of-bound access
    if (row < num_rows && col < num_cols) {
        float f = feats[row * ldf + col];
        feats_sq[row * lds + col] = f * f;
    }
}