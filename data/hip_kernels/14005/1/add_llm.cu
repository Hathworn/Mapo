#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C"
__global__ void add(int n, float *a, float *b, float *sum)
{
    // Cache blockDim.x for potential optimization
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for better parallelization
    for (int i = idx; i < n; i += blockDim.x * gridDim.x)
    {
        sum[i] = a[i] + b[i];
    }
}