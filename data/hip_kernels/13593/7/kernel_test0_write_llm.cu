#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    unsigned int* orig_ptr = (unsigned int*) (_ptr + blockIdx.x*BLOCKSIZE);
    unsigned int* ptr = orig_ptr;

    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);
    unsigned int pattern = 1;
    unsigned long mask = 4;

    *ptr = pattern;

    while (ptr < block_end) {
        ptr = (unsigned int*)((unsigned long)orig_ptr | mask);
        if (ptr >= block_end) {  // Break if beyond block boundary
            break;
        }
        
        *ptr = pattern;  // Write pattern

        pattern <<= 1;  // Optimize pattern update using compound assignment
        mask <<= 1;     // Optimize mask update using compound assignment

        // Skip redundant mask shifting when ptr equals orig_ptr
        if (ptr == orig_ptr) {
            continue;
        }
    }
}