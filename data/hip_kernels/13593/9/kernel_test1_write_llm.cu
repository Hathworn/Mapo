#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    unsigned int tid = threadIdx.x; // Use threadIdx for fine-grained parallelism
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x * BLOCKSIZE);

    if (ptr < (unsigned long*) end_ptr && tid < BLOCKSIZE / sizeof(unsigned long)) {
        ptr[tid] = (unsigned long) &ptr[tid]; // Assign value using individual threads
    }
}