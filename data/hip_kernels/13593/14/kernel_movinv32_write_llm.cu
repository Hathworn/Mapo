#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_movinv32_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int lb, unsigned int sval, unsigned int offset)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int thread_id = threadIdx.x;

    // Exit early if block's starting address is beyond end_ptr
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }
    
    unsigned int k = offset + thread_id;  // Offset calculation for each thread
    unsigned int pat = pattern;
    
    // Precompute the number of elements per thread
    unsigned int elements_per_thread = BLOCKSIZE / sizeof(unsigned int) / blockDim.x;
    
    for (unsigned int i = 0; i < elements_per_thread; i++){
        unsigned int index = thread_id * elements_per_thread + i;
        ptr[index] = pat;
        k++;
        if (k >= 32){
            k = 0;
            pat = lb;
        }else{
            pat = pat << 1;
            pat |= sval;
        }
    }

    return;
}