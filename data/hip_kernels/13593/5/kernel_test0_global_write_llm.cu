#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int* orig_ptr = ptr;

    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Write initial pattern
    *ptr = pattern;

    // Loop through memory until the end pointer is reached
    while(ptr < end_ptr) {
        ptr = (unsigned int*) (((unsigned long)orig_ptr) | mask); // Calculate next pointer
        if (ptr == orig_ptr) {
            mask = mask << 1; // Shift mask if there is no change to the pointer
            continue;
        }
        if (ptr >= end_ptr) {
            break; // Exit loop if the pointer has reached the end
        }

        *ptr = pattern; // Write pattern to calculated pointer
        pattern = pattern << 1; // Shift pattern
        mask = mask << 1; // Shift mask for next iteration
    }
    return;
}