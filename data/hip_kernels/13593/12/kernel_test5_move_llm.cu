#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check boundary condition
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Calculate half of the count to be processed
    unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Use shared memory for faster data transfer
    __shared__ unsigned int shared_mem[BLOCKSIZE / sizeof(unsigned int)];

    // Load data into shared memory
    unsigned int idx = threadIdx.x;
    if (idx < half_count) {
        shared_mem[idx] = ptr[idx];
        __syncthreads();

        // Copy second half from shared memory
        ptr_mid[idx] = shared_mem[idx];
        __syncthreads();

        // Perform main operations using shared memory
        if (idx < half_count - 8) {
            shared_mem[idx + 8] = shared_mem[idx];
        }
        if (idx < 8) {
            shared_mem[idx] = shared_mem[half_count - 8 + idx];
        }
        __syncthreads();

        // Write results back to global memory
        if (idx < half_count - 8) {
            ptr[idx + 8] = shared_mem[idx + 8];
        }
        if (idx < 8) {
            ptr[idx] = shared_mem[idx];
        }
    }
}