#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    // Calculate the global index
    unsigned int idx = blockIdx.x * BLOCKSIZE / sizeof(unsigned int);
    unsigned int* ptr = (unsigned int*)_ptr + idx;

    // Check bounds just once using current thread's workload
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    unsigned int limit = BLOCKSIZE / sizeof(unsigned int);

    // Unroll the loop for efficiency, assuming MOD_SZ is known at compile time
    for (unsigned int i = offset; i < limit; i += MOD_SZ) {
        ptr[i] = p1; // Set p1 for all offset matched positions
    }

    for (unsigned int i = 0; i < limit; i++) {
        // Use recalculated idx for correct indexing and set p2 for others
        if (i % MOD_SZ != offset) {
            ptr[i] = p2;
        }
    }
}