#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_init(char* _ptr, char* end_ptr)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check if the pointer is out of bounds early
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    unsigned int p1 = 1;
    // Unroll the loop manually for better performance
    for (int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i += 16) {
        unsigned int p2 = ~p1;

        // Use a loop to reduce repetitive code
        for (int j = 0; j < 16; j += 4) {
            ptr[i + j] = p1;
            ptr[i + j + 1] = p1;
            ptr[i + j + 2] = p2;
            ptr[i + j + 3] = p2;
        }

        p1 <<= 1; // Use compound assignment for clarity
        p1 = (p1 == 0) ? 1 : p1; // Simplify p1 reset condition
    }
}