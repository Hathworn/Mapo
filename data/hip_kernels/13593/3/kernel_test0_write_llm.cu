#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    // Calculate global index directly
    unsigned int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int* ptr = (unsigned int*)(_ptr + globalIdx * sizeof(unsigned int));

    // Check bounds condition
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Write pattern to memory
    *ptr = pattern;
}