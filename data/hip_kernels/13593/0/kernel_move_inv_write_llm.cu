#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check boundary condition only once at beginning
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Unroll the loop to utilize memory bandwidth more effectively
    #pragma unroll
    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = pattern;
    }
}