#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* start_ptr = (unsigned int*) _start_ptr;

    // Optimize the bounds check by computing end_ptr inside the loop
    if (ptr < (unsigned int*) end_ptr) {
        int tid = threadIdx.x;
        int elements = BLOCKSIZE / sizeof(unsigned int);

        // Optimize using a loop with thread parallelism
        for (int i = tid; i < elements; i += blockDim.x) {
            ptr[i] = start_ptr[i];
        }
    }
}