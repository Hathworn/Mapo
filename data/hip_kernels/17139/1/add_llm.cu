#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c, int N)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure that each thread operates within the valid data range
    if (tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}