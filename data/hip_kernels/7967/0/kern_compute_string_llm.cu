#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define MEMSIZE 30

/* Function computing the final string to print */
__global__ void kern_compute_string(char *res, char *a, char *b, char *c, int length)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index

    if (i < length)
    {
        res[i] = a[i] + b[i] + c[i]; // Perform computations on in-bounds index
    }
}