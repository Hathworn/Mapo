#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for remap reduction
__global__ void remap_reduction(uint32_t *d_reduction, uint32_t *d_mapping, uint32_t *old_d_ij_buf, uint32_t sum_prev_size, uint32_t prev_size, uint32_t *new_d_ij_buf, uint32_t new_size)
{
    uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;

    // Ensure thread is within bounds
    if (t_index < prev_size) {
        // Check if current reduction element is active
        if (d_reduction[t_index]) {
            uint32_t index = d_mapping[t_index];
            uint32_t offset = 2 * sum_prev_size + t_index;
            uint32_t i = old_d_ij_buf[offset];
            uint32_t j = old_d_ij_buf[offset + prev_size];

            // Sort pairs if needed when sum_prev_size is zero
            if (!sum_prev_size && j < i) {
                uint32_t temp = i;
                i = j;
                j = temp;
            }

            // Remap to new buffer
            new_d_ij_buf[2 * sum_prev_size + index] = i;
            new_d_ij_buf[2 * sum_prev_size + new_size + index] = j;
        }
    }
}