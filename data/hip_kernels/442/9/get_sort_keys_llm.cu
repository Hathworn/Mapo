#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void get_sort_keys(unsigned char* d_hashes, uint32_t *d_sort_keys, uint32_t *d_sort_indices, uint32_t r, uint32_t num_keys)
{
    uint32_t index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < num_keys) {
        d_sort_indices[index] = index;

        // Use a single pointer arithmetic operation for efficiency
        unsigned char* input = d_hashes + index * 30 + 3 * r;
        uint32_t sort_key = (static_cast<uint32_t>(input[0]) << 16) | (static_cast<uint32_t>(input[1]) << 8) | static_cast<uint32_t>(input[2]);
        d_sort_keys[index] = sort_key;
    }
}