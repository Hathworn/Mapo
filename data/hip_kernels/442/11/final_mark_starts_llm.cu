#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void final_mark_starts(uint8_t *hashes, uint32_t *sort_indices, uint32_t *off_map, uint32_t r, uint32_t hash_count)
{
    uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;
    if (t_index < hash_count) {
        // Optimize calculation of previous index using conditional operator for efficiency
        uint32_t t_prev_index = t_index == 0 ? hash_count - 1 : t_index - 1;

        uint32_t index = sort_indices[t_index];
        uint32_t prev_index = sort_indices[t_prev_index];

        // Pre-calculate hash offset, removing multiplication with sizeof(unsigned char) as it's 1
        unsigned char* hash = hashes + index * 30 + r * 3;
        unsigned char* prev_hash = hashes + prev_index * 30 + r * 3;

        // Unroll hash key calculation for potential compiler optimization
        uint64_t key = ((uint64_t)hash[0]) << 40 | ((uint64_t)hash[1]) << 32 | ((uint64_t)hash[2]) << 24
                     | ((uint64_t)hash[3]) << 16 | ((uint64_t)hash[4]) << 8 | hash[5];

        uint64_t prev_key = ((uint64_t)prev_hash[0]) << 40 | ((uint64_t)prev_hash[1]) << 32 | ((uint64_t)prev_hash[2]) << 24
                          | ((uint64_t)prev_hash[3]) << 16 | ((uint64_t)prev_hash[4]) << 8 | prev_hash[5];

        // Use direct assignment with a conditional check to minimize operations
        off_map[t_index] = (key ^ prev_key) != 0 ? 1 : 0;
    }
}