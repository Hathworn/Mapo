#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void map_index_to_prefix( uint8_t *hashes, uint32_t *sort_indices, uint32_t *comb_count, uint32_t *comb_sum, uint32_t *comb_prefix, uint32_t r, uint32_t size)
{
    uint32_t t_index = blockDim.x * blockIdx.x + threadIdx.x;
    if(t_index < size) {
        // Load frequently used values into registers
        uint32_t index = sort_indices[t_index];
        unsigned char* hash = hashes + index * 30 + r * 3;
        uint32_t key = (hash[0] << 16) | (hash[1] << 8) | hash[2];
        
        uint32_t count = comb_count[key];
        uint32_t sum = comb_sum[key];
        
        // Use shared memory for comb_prefix to reduce memory latency
        __shared__ uint32_t shared_comb_prefix[BLOCK_SIZE];
        for(int i = threadIdx.x; i < (sum - count); i += blockDim.x) {
            shared_comb_prefix[i] = key;
        }
        
        // Write back to global memory
        for(int i = sum - count + threadIdx.x; i < sum; i += blockDim.x) {
            comb_prefix[i] = shared_comb_prefix[i - (sum - count)];
        }
    }
}