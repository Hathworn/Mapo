#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void invert(float *output, int* input, const int size)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Use shared memory for better performance if input size is large
    extern __shared__ int shared_input[];

    if (i < size) 
    {
        // Load data to shared memory
        shared_input[threadIdx.x] = input[i];
        __syncthreads();

        int d = shared_input[threadIdx.x];
        if (d > 0)
        {
            output[i] = __fdividef(1.f, d);
        }
    }
}