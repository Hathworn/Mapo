#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addOneRowPerThread(double* a, double* b, double* c, int n)
{
    // Calculate the row for current thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Unrolling to reduce loop overhead and improving memory coalescing
    if (row < n)
    {
        int idx = row * n;
        int i = 0;

        #pragma unroll
        for (; i <= n - 4; i += 4)
        {
            c[idx + i]     = a[idx + i]     + b[idx + i];
            c[idx + i + 1] = a[idx + i + 1] + b[idx + i + 1];
            c[idx + i + 2] = a[idx + i + 2] + b[idx + i + 2];
            c[idx + i + 3] = a[idx + i + 3] + b[idx + i + 3];
        }

        // Handle the rest of the elements
        for (; i < n; i++)
        {
            c[idx + i] = a[idx + i] + b[idx + i];
        }
    }
}