#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_inv(const int* __restrict__ destination_offsets, const int* __restrict__ source_indices, const float* __restrict__ out_degrees, const int node_count, const float* __restrict__ input, float* __restrict__ output)
{
    int dest = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (dest < node_count)
    {
        int srcStart = destination_offsets[dest];
        int srcEnd = destination_offsets[dest + 1];
        int in_degree = srcEnd - srcStart;
        float rank = 0.0f;

        // Use a local variable for DECAY to avoid multiple loads
        const float decay_local = DECAY;

        // Optimize by checking in-degree once
        if (in_degree > 0)
        {
            #pragma unroll
            for (int srcIdx = srcStart; srcIdx < srcEnd; ++srcIdx)
            {
                int src = source_indices[srcIdx];
                // Minimize memory fetch and repeat calculations
                float input_src = input[src];
                float out_degree_src = out_degrees[src];
                float contrib = (input_src * decay_local) * out_degree_src;
                rank += contrib;
            }
        }
        
        // Calculate rank with a precomputed constant part
        output[dest] = rank + (1.0f - decay_local);
    }
}