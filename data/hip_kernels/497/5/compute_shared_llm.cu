#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_shared(const int* destination_offsets, const int* source_indices, const int* out_degrees, const int node_count, const float* input, float *output)
{
    int dest = blockDim.x*blockIdx.x + threadIdx.x;
    __shared__ int s_dest_off[BLOCK_SIZE + 1];

    // Check if destination is within bounds
    if (dest < node_count)
    {
        // Load destination offsets into shared memory
        s_dest_off[threadIdx.x] = destination_offsets[dest];
        if (threadIdx.x == BLOCK_SIZE - 1 || dest == node_count - 1) {
            s_dest_off[threadIdx.x + 1] = destination_offsets[dest + 1];
        }
        __syncthreads();

        int srcStart = s_dest_off[threadIdx.x];
        int srcEnd = s_dest_off[threadIdx.x + 1];
        float rank = 0.0f;

        // Optimize by unrolling loop if possible
        if (srcStart < srcEnd) {
            for (int srcIdx = srcStart; srcIdx < srcEnd; ++srcIdx) {
                int src = source_indices[srcIdx];
                float contrib = ((input[src] * DECAY) / out_degrees[src]);
                rank += contrib;
            }
        }
        // Update the output value
        output[dest] = rank + (1.0f - DECAY);
    }
}