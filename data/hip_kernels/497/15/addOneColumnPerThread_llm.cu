#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addOneColumnPerThread(double* a, double* b, double* c, int n)
{
    // Get the column for current thread
    int column = (blockIdx.x * blockDim.x + threadIdx.x);

    // Ensure thread does useful work
    if (column < n)
    {
        int offset = column;  // Precompute offset
        int stride = n;       // Avoid recalculating stride
        for (int i = 0; i < n; i++)
        {
            // Use precomputed values for better performance
            c[i * stride + offset] = a[i * stride + offset] + b[i * stride + offset];
        }
    }
}