#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addOneElementPerThread(double* a, double* b, double* c, int n)
{
    // Calculate global thread ID for both x and y dimensions
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Calculate the linear index
    int id = idy * n + idx;

    // Ensure within bounds before operation
    if (idx < n && idy < n)
    {
        // Efficiently load values from global memory, compute, and store result
        double val_a = a[id];
        double val_b = b[id];
        c[id] = val_a + val_b;
    }
}