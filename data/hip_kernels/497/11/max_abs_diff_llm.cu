#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void max_abs_diff(float* diff, const float* output1, const float* output2, const int size)
{
    extern __shared__ float s_max[];
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int tx = threadIdx.x;

    // Load shared memory only for valid threads within bounds
    if (i < size)
    {
        float o1 = output1[i];
        s_max[tx] = (o1 == -1) ? -1.0f : fabsf(o1 - output2[i]);
    }
    else
    {
        s_max[tx] = -1.0f;
    }

    __syncthreads();  // Ensure shared memory is fully populated

    // Reduce within shared memory
    for (int j = blockDim.x / 2; j > 0; j >>= 1)
    {
        if (tx < j)
        {
            s_max[tx] = fmaxf(s_max[tx], s_max[tx + j]);
        }
        __syncthreads();  // Synchronize at each step of reduction
    }

    // Write block result to global memory
    if (tx == 0)
    {
        diff[blockIdx.x] = s_max[0];
    }
}