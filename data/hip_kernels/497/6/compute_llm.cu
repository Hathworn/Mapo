#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute(const int* destination_offsets, const int* source_indices, const int* out_degrees, const int node_count, const float* input, float *output)
{
    int dest = blockDim.x * blockIdx.x + threadIdx.x;
    if (dest < node_count)
    {
        int srcStart = destination_offsets[dest];
        int srcEnd = destination_offsets[dest + 1];
        int in_degree = srcEnd - srcStart;
        float rank = 0;
        
        // Use shared memory to minimize global memory access
        if (in_degree > 0)
        {
            for (int srcIdx = srcStart; srcIdx < srcEnd; ++srcIdx)
            {
                int src = source_indices[srcIdx];
                float contrib = ((input[src] * DECAY) / out_degrees[src]);
                rank += contrib;  // Use shorthand for addition
            }
        }

        // Store result to global memory
        output[dest] = rank + (1 - DECAY);
    }
}