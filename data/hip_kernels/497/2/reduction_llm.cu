#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduction(float *g_data, int n)
{
    __shared__ float partialSum[NUM_ELEMENTS];
    unsigned int t = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + t;  // Calculate the global index
    if (idx < n) partialSum[t] = g_data[idx]; // Load only if index is within array bounds
    else partialSum[t] = 0.0f;  // Initialize to zero if out of bounds
    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i >>= 1)
    {
        if (t < i && idx + i < n)  // Ensure access is within bounds
        {
            partialSum[t] += partialSum[t + i];
        }
        __syncthreads();
    }
    if (t == 0)
    {
        g_data[blockIdx.x] = partialSum[0];  // Store the result in the block's starting location
    }
}