#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void TestDimensions()
{
    // Calculate number of threads per block and total blocks (unchanged)
    int const num_threads_per_block = blockDim.x * blockDim.y * blockDim.z;
    int const num_blocks = gridDim.x * gridDim.y * gridDim.z;
    int const threads_stride = num_threads_per_block * num_blocks;

    // Optimize calculation of thread_id using a more concise formula
    int const thread_id = ((blockIdx.z * gridDim.y + blockIdx.y) * gridDim.x + blockIdx.x) * num_threads_per_block +
                          threadIdx.z * blockDim.y * blockDim.x +
                          threadIdx.y * blockDim.x +
                          threadIdx.x;

    // Print once for the first thread
    if( thread_id == 0 ) 
    {
        printf( "gridDim   = x: %6d / y: %6d / z: %6d\r\n",
                gridDim.x, gridDim.y, gridDim.z );
        printf( "blockDim  = x: %6d / y: %6d / z: %6d\r\n",
                blockDim.x, blockDim.y, blockDim.z );
        printf( "num_threads_per_block: %6d\r\n", num_threads_per_block );
        printf( "num_blocks           : %6d\r\n", num_blocks );
        printf( "threads_stride       : %6d\r\n", threads_stride );
    }

    // Print details for all threads
    printf( "tidx | %6d | %6d | %6d | bidx | %6d | %6d | %6d | "
            "gdim | %6d | %6d | %6d | bdim | %6d | %6d | %6d | "
            "thread_id | %6d |\r\n",
            threadIdx.x, threadIdx.y, threadIdx.z,
            blockIdx.x, blockIdx.y, blockIdx.z,
            gridDim.x, gridDim.y, gridDim.z,
            blockDim.x, blockDim.y, blockDim.z, thread_id );

    return;
}