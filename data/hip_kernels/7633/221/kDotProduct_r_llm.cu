#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Efficient handling of boundary condition
    for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
        sum += a[eidx] * b[eidx];
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();
    
    // Unroll loop to reduce execution divergence
    if (threadIdx.x < 128) { shmem[threadIdx.x] += shmem[threadIdx.x + 128]; }
    __syncthreads();
    if (threadIdx.x < 64) { shmem[threadIdx.x] += shmem[threadIdx.x + 64]; }
    __syncthreads();

    // Further optimization for warp-level execution
    if (threadIdx.x < 32) {
        volatile float* mysh = &shmem[threadIdx.x];
        *mysh += mysh[32];
        *mysh += mysh[16];
        *mysh += mysh[8];
        *mysh += mysh[4];
        *mysh += mysh[2];
        *mysh += mysh[1];
    }

    // Only write result by first thread in block
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}