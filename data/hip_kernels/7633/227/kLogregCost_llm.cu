#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * blockDim.x + threadIdx.x;  // Simplifying thread index calculation

    if (tx < numCases) {
        const int label = __float2int_rn(labels[tx]);  // Faster integer conversion
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        if (labelp != maxp) {
            correctProbs[tx] = 0.0f;  // Use 0.0f for explicit float
        } else {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp);  // Parentheses for clear logic evaluation
            }
            correctProbs[tx] = 1.0f / static_cast<float>(numMax);  // Consistent float casting
        }
    }
}