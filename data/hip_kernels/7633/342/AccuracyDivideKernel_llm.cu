#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
  // Use local variable to reduce memory access latency
  float acc_local = *accuracy;
  acc_local /= N;
  *accuracy = acc_local;
}