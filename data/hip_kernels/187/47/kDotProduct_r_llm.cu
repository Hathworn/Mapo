#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;  // Use a local accumulator for reduction

    if (eidx < numElements) { // Ensure index within bounds
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            sum += a[eidx] * b[eidx];
        }
    }

    shmem[threadIdx.x] = sum; // Store local sum into shared memory
    __syncthreads();

    // Perform block-wise reduction using efficient warp shuffle
    for (int offset = DP_BLOCKSIZE >> 1; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            shmem[threadIdx.x] += shmem[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Write the reduced result for this block to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}