#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Compute a unique thread index using block and thread indices
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    
    // Unrolling loop to reduce overhead and improve performance
    #pragma unroll
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Calculate positions in the target and source arrays
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Copy from source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}