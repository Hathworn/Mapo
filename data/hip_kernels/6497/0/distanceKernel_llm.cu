#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float distance(float x1, float x2)
{
    return sqrtf((x2 - x1) * (x2 - x1)); // Use sqrtf for single precision
}

__device__ float scale(int i, int n)
{
    return ((float)i) / (n - 1);
}

__global__ void distanceKernel(float *d_out, float ref, int len)
{
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < len) // Add boundary check
    {
        const float x = scale(i, len);
        d_out[i] = distance(x, ref);
        printf("i = %2d: dist from %f to %f is %f.\n", i, ref, x, d_out[i]);
    }
}