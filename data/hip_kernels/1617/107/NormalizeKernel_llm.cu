#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Use blockIdx.y and blockDim.y for better readability
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Use blockIdx.x and blockDim.x for better readability

    if (i < h && j < w) // Use logical 'and' for better clarity
    {
        const int pos = i * s + j;

        float scale = normalization_factor[pos];

        float invScale = (scale != 0.0f) ? (1.0f / scale) : 1.0f; // Use '!=' for better readability

        image[pos] *= invScale;
    }
}