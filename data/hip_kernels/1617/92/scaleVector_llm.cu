#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    // Calculate the global thread ID
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize branch by using a single condition check
    if (pos < len) 
    {
        d_res[pos] = d_src[pos] * scale;
    }
}