#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Precompute scaled flow values
    float scaled_u = u[flow_row_offset + j] * time_scale;
    float scaled_v = v[flow_row_offset + j] * time_scale;
    
    // Compute pixel coordinates directly
    float cx = scaled_u + (float)j + 1.0f;
    float cy = scaled_v + (float)i + 1.0f;

    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    // Calculate value once
    float value = src[image_row_offset + j];
    
    int tx = (int) px;
    int ty = (int) py;
    
    float weight;

    // Using a helper lambda for bounds check and atomic operations
    auto atomicAddSafe = [&] (int x, int y, float val, float wt) {
        if (!(x >= w || x < 0 || y >= h || y < 0)) {
            _atomicAdd(dst + y * image_stride + x, val * wt);
            _atomicAdd(normalization_factor + y * image_stride + x, wt);
        }
    };
    
    // Fill pixel containing bottom right corner
    weight = dx * dy;
    atomicAddSafe(tx, ty, value, weight);

    // Fill pixel containing bottom left corner
    tx -= 1;
    weight = (1.0f - dx) * dy;
    atomicAddSafe(tx, ty, value, weight);
    
    // Fill pixel containing upper left corner
    ty -= 1;
    weight = (1.0f - dx) * (1.0f - dy);
    atomicAddSafe(tx, ty, value, weight);
    
    // Fill pixel containing upper right corner
    tx += 1;
    weight = dx * (1.0f - dy);
    atomicAddSafe(tx, ty, value, weight);
}