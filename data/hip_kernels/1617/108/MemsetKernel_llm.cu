#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val) {
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float* image) {
    // Optimize memory accesses by using blockDim.y and blockDim.x directly
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory when possible or reduce register pressure
    if (i < h && j < w) {
        const int pos = i * w + j;
        image[pos] = value;
    }
}