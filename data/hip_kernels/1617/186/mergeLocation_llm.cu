#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread within bounds and calculate scaled location in one step
    if (ptidx < npoints)
    {
        const short2 loc = loc_[ptidx];
        const float scaled_x = loc.x * scale;
        const float scaled_y = loc.y * scale;

        x[ptidx] = scaled_x;
        y[ptidx] = scaled_y;
    }
}