#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GradientAverageKernel(float4 *D, float4 *TD, unsigned int *NEIGHBOR, unsigned int *NBOFFSETS, unsigned int *nNeighbors, unsigned int nVertices)
{
    int vidxb = 4 * (blockIdx.x * blockDim.x) + threadIdx.x;
    int vidx;
    float4 nbd, td;
    
    // Use shared memory to cache vertices data 
    __shared__ float4 SI[4 * BLOCK_SIZE_AVGG];
    
    int bidx = 4 * threadIdx.x;

    // Load vertices into shared memory (efficient memory access)
    for (vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_AVGG; vidx += BLOCK_SIZE_AVGG) {
        if (vidx < nVertices) {
            SI[bidx] = D[vidx];
            bidx++;
        }
    }

    __syncthreads();
    bidx = 4 * threadIdx.x;

    // Precompute vertex gradients using cached vertices
    for (vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_AVGG; vidx += BLOCK_SIZE_AVGG) {
        if (vidx < nVertices) {
            int offset = NBOFFSETS[vidx];
            int N = nNeighbors[vidx];
            
            td = SI[bidx++];
            
            for (int n = 0; n < N; n++) {
                int soffset = NEIGHBOR[offset + n];
                
                if (soffset >= vidxb && soffset < vidxb + 4 * BLOCK_SIZE_AVGG) { // Efficient shared memory access
                    nbd = SI[soffset - vidxb];
                } else {
                    nbd = D[soffset];
                }
                
                td.x += nbd.x;
                td.y += nbd.y;
                td.z += nbd.z;
            }
            
            td.x /= (float)(N + 1);
            td.y /= (float)(N + 1);
            td.z /= (float)(N + 1);
            
            TD[vidx] = td;
        }
    }
}