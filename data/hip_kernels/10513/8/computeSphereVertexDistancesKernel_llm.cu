#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeSphereVertexDistancesKernel(float4 *V, float *dist, unsigned int *NEIGHBOR, unsigned int *NBOFFSETS, unsigned int *nNeighbors, unsigned int nVertices, float circumference)
{
    int n, N;
    int offset, soffset;

    // Calculate a unique index for each thread
    int vidxb = 4 * (blockIdx.x * blockDim.x) + threadIdx.x;

    int basevert = 4 * (blockIdx.x * blockDim.x);

    float4 nv, tv;
    float dot, n1, n2, norm;

    // Cache shared memory for vertex data
    __shared__ float4 SI[4 * BLOCK_SIZE_CVD];

    // Use a stride pattern to load vertex data into shared memory
    for (int vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_CVD; vidx += BLOCK_SIZE_CVD) {
        if (vidx < nVertices) {
            SI[threadIdx.x] = V[vidx];
            threadIdx.x += BLOCK_SIZE_CVD;
        }
    }

    __syncthreads();

    // Separate block to reduce warp divergence
    for (int vidx = vidxb; vidx < vidxb + 4 * BLOCK_SIZE_CVD; vidx += BLOCK_SIZE_CVD) {
        if (vidx < nVertices) {
            offset = NBOFFSETS[vidx];
            N = nNeighbors[vidx];
            tv = SI[threadIdx.x];

            for (n = 0; n < N; n++) {
                soffset = NEIGHBOR[offset + n];

                // Determine if neighbor is in cache; otherwise, fetch from global
                float4 nv = (int tab = soffset - basevert; (tab > 0 && tab < 4 * BLOCK_SIZE_CVD)) ? SI[tab] : V[soffset];

                // Calculate dot products and norms using intrinsic functions
                dot = __fmul_rn(tv.x, nv.x);
                dot = __fadd_rn(dot, __fmul_rn(tv.y, nv.y));
                dot = __fadd_rn(dot, __fmul_rn(tv.z, nv.z));

                n1 = __fmul_rn(tv.x, tv.x);
                n1 = __fadd_rn(n1, __fmul_rn(tv.y, tv.y));
                n1 = __fadd_rn(n1, __fmul_rn(tv.z, tv.z));

                n2 = __fmul_rn(nv.x, nv.x);
                n2 = __fadd_rn(n2, __fmul_rn(nv.y, nv.y));
                n2 = __fadd_rn(n2, __fmul_rn(nv.z, nv.z));

                norm = __fmul_rn(__fsqrt_rn(n1), __fsqrt_rn(n2));

                // Handle edge cases with direct assignments
                if (norm < 1.0e-7f) {
                    dist[offset + n] = 0.0f;
                } else if (fabsf(dot) > norm) {
                    dist[offset + n] = 0.0f;
                } else {
                    dist[offset + n] = __fmul_rn(circumference, fabsf(acosf(dot / norm)));
                }
            }
        }
    }
}