#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateGradientsKernel(float4 *D, float4 *TD, unsigned int nVertices)
{
    // Calculate global thread index
    int localIdx = threadIdx.x;
    int vidx = 4 * (blockIdx.x * blockDim.x) + localIdx; 
    
    // Unroll the loop for better performance
    D[vidx] = TD[vidx];
    D[vidx + BLOCK_SIZE_AVGG] = TD[vidx + BLOCK_SIZE_AVGG];
    D[vidx + 2 * BLOCK_SIZE_AVGG] = TD[vidx + 2 * BLOCK_SIZE_AVGG];
    D[vidx + 3 * BLOCK_SIZE_AVGG] = TD[vidx + 3 * BLOCK_SIZE_AVGG];
}