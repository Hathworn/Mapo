#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void kernel_test0_write(char* _ptr, char* end_ptr, unsigned int pattern, unsigned int* err, unsigned long* err_addr, unsigned long* err_expect, unsigned long* err_current, unsigned long* err_second_read)
{
    // Calculate the starting index based on threadIdx and blockIdx
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Avoid accessing out-of-bounds memory
    if (ptr + idx * blockDim.x >= (unsigned int*) end_ptr) {
        return;
    }

    // Use each thread to handle a part of the workload
    if (idx < BLOCKSIZE / sizeof(unsigned int)) {
        ptr[idx] = pattern;
    }
}