#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_init(char* _ptr, char* end_ptr)
{
    // Calculate global index
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);

    // Check if the pointer exceeds end_ptr
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Initialize p1 and precompute repeated pattern
    unsigned int p1 = 1;
    unsigned int pattern[16];

    for (unsigned int i = 0; i < BLOCKSIZE / sizeof(unsigned int); i += 16) {
        unsigned int p2 = ~p1;

        // Vectorize pattern initialization
        pattern[0] = p1;       pattern[1] = p1;
        pattern[2] = p2;       pattern[3] = p2;
        pattern[4] = p1;       pattern[5] = p1;
        pattern[6] = p2;       pattern[7] = p2;
        pattern[8] = p1;       pattern[9] = p1;
        pattern[10] = p2;      pattern[11] = p2;
        pattern[12] = p1;      pattern[13] = p1;
        pattern[14] = p2;      pattern[15] = p2;

        // Unroll the loop by assigning entire pattern at once
        for (unsigned int j = 0; j < 16; ++j) {
            ptr[i + j] = pattern[j];
        }

        p1 = p1 << 1;
        if (p1 == 0) {
            p1 = 1;
        }
    }

    return;
}