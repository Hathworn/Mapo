#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void kernel_test0_global_write(char* _ptr, char* _end_ptr)
{
    unsigned int* ptr = (unsigned int*)_ptr;
    unsigned int* end_ptr = (unsigned int*)_end_ptr;
    unsigned int* orig_ptr = ptr;

    unsigned int pattern = 1;
    unsigned long mask = 4;

    *ptr = pattern;

    while(ptr < end_ptr){
        ptr = (unsigned int*)(((unsigned long)orig_ptr) | mask);
        
        // Avoid redundant calculation when ptr equals orig_ptr
        if (ptr >= end_ptr || ptr == orig_ptr){
            mask = mask << 1;
            continue;
        }

        *ptr = pattern;
        
        // Use a consistent left shift for both pattern and mask
        pattern <<= 1;
        mask <<= 1;
    }
}