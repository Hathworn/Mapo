#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_move_inv_write(char* _ptr, char* end_ptr, unsigned int pattern)
{
    // Calculate the current thread's index in the grid
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int* ptr = (unsigned int*) (_ptr + idx * sizeof(unsigned int));

    // Terminate threads with indices beyond the end pointer
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Write the pattern to memory
    *ptr = pattern;
}