#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_modtest_write(char* _ptr, char* end_ptr, unsigned int offset, unsigned int p1, unsigned int p2)
{
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    // Early exit if the pointer is beyond the end
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Use a single loop with conditional logic to reduce branching
    unsigned int idx = threadIdx.x; // Use thread index to parallelize
    unsigned int step_size = BLOCKSIZE / sizeof(unsigned int);
    unsigned int limit = min(step_size, MOD_SZ); // Ensure we don't overrun

    for (unsigned int i = idx; i < step_size; i += blockDim.x) {
        ptr[i] = (i % MOD_SZ == offset) ? p1 : p2;
    }
}