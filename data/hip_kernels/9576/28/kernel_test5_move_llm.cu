#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test5_move(char* _ptr, char* end_ptr)
{
    // Calculate the initial offset for each block
    unsigned int* ptr = reinterpret_cast<unsigned int*>(_ptr + blockIdx.x * BLOCKSIZE);
    
    // If the pointer exceeds end_ptr, exit the kernel
    if (ptr >= reinterpret_cast<unsigned int*>(end_ptr)) {
        return;
    }

    // Calculate the half block count in terms of unsigned int
    unsigned int half_count = BLOCKSIZE / sizeof(unsigned int) / 2;
    unsigned int* ptr_mid = ptr + half_count;

    // Use shared memory to reduce global memory access
    __shared__ unsigned int shared_mem[BLOCKSIZE / sizeof(unsigned int)];

    // Move first half to shared memory
    for (unsigned int i = threadIdx.x; i < half_count; i += blockDim.x) {
        shared_mem[i] = ptr[i];
    }
    __syncthreads();

    // Move data from shared memory to ptr_mid
    for (unsigned int i = threadIdx.x; i < half_count; i += blockDim.x) {
        ptr_mid[i] = shared_mem[i];
    }
    __syncthreads();

    // Reorganize ptr using shared memory
    for (unsigned int i = threadIdx.x; i < half_count - 8; i += blockDim.x) {
        ptr[i + 8] = ptr_mid[i];
    }
    __syncthreads();

    for (unsigned int i = threadIdx.x; i < 8; i += blockDim.x) {
        ptr[i] = ptr_mid[half_count - 8 + i];
    }
}