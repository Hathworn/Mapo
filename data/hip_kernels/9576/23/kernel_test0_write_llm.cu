#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test0_write(char* _ptr, char* end_ptr)
{
    // Calculate original pointer and check bounds
    unsigned int* orig_ptr = (unsigned int*)(_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* ptr = orig_ptr;
    if (ptr >= (unsigned int*)end_ptr) {
        return;
    }

    // Calculate the end of the block
    unsigned int* block_end = orig_ptr + BLOCKSIZE / sizeof(unsigned int);

    unsigned int pattern = 1;
    unsigned long mask = 4;

    // Write initial pattern
    *ptr = pattern;

    while (ptr < block_end) {
        // Use bit manipulation to calculate new pointer efficiently
        ptr = (unsigned int*)(((unsigned long)orig_ptr) | mask);

        // Avoid overwriting the original pointer
        if (ptr == orig_ptr) {
            mask = mask << 1;
            continue;
        }

        // Break if the pointer exceeds block boundary
        if (ptr >= block_end) {
            break;
        }

        // Write pattern
        *ptr = pattern;

        // Shift pattern and mask for next iteration
        pattern <<= 1;
        mask <<= 1;
    }
}
```
