#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_test7_write(char* _ptr, char* end_ptr, char* _start_ptr, unsigned int* err)
{
    unsigned int i;
    unsigned int* ptr = (unsigned int*) (_ptr + blockIdx.x * BLOCKSIZE);
    unsigned int* start_ptr = (unsigned int*) _start_ptr;

    // Return early if the current block exceeds the end pointer
    if (ptr >= (unsigned int*) end_ptr) {
        return;
    }

    // Use loop unrolling to optimize memory writes
    #pragma unroll
    for (i = 0; i < BLOCKSIZE / sizeof(unsigned int); i++) {
        ptr[i] = start_ptr[i];
    }

    return;
}