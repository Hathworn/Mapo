#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_test1_write(char* _ptr, char* end_ptr, unsigned int* err)
{
    unsigned long* ptr = (unsigned long*) (_ptr + blockIdx.x*BLOCKSIZE);

    // Ensure memory access is within bounds
    if (ptr >= (unsigned long*) end_ptr) {
        return;
    }

    // Use `threadIdx.x` to parallelize the loop effectively
    int idx = threadIdx.x;
    if (idx < BLOCKSIZE / sizeof(unsigned long)) {
        ptr[idx] = (unsigned long) &ptr[idx];
    }

    return;
}