#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void mprts_update_offsets(int nr_total_blocks, uint *d_off, uint *d_spine_sums)
{
    // Load computation for blocks in shared memory
    int bid = threadIdx.x + THREADS_PER_BLOCK * blockIdx.x;
    
    if (bid < nr_total_blocks) { // Use less-than to avoid accessing out-of-bounds
        // Coalesced memory access
        d_off[bid] = __ldg(&d_spine_sums[bid * CUDA_BND_STRIDE]);
    }
}