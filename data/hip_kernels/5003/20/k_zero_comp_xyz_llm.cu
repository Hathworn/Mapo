#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void k_zero_comp_xyz(float *data, uint n, uint stride)
{
    // Calculate global index for 2D grid
    uint globalIdx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (globalIdx < n) {
        // Calculate the offset for the second dimension
        data[globalIdx + blockIdx.y * stride] = 0.f;
    }
}