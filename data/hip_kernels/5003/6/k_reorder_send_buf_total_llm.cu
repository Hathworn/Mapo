#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void k_reorder_send_buf_total(int nr_prts, int nr_total_blocks, uint *d_bidx, uint *d_sums, float4 *d_xi4, float4 *d_pxi4, float4 *d_xchg_xi4, float4 *d_xchg_pxi4)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= nr_prts) return;

    // Use a register to cache frequently used data
    uint bidx = d_bidx[i];
    
    // Optimize by removing redundant global memory accesses
    if (bidx == CUDA_BND_S_OOB) {
        int j = d_sums[i];
        
        // Coalesced memory writes
        float4 xi = d_xi4[i];
        float4 pxi = d_pxi4[i];
        
        d_xchg_xi4[j]  = xi;
        d_xchg_pxi4[j] = pxi;
    }
}