#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void k_count_received(int nr_total_blocks, uint *d_n_recv_by_block, uint *d_spine_cnts)
{
    int bid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bid is within bounds before proceeding
    if (bid >= nr_total_blocks) return;

    // Load and store using coalesced memory access
    uint recv_count = d_n_recv_by_block[bid];
    d_spine_cnts[bid * 10 + CUDA_BND_S_NEW] = recv_count;
}