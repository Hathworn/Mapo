#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int n, float *x, float *y) {
    
    // Calculate the global index of each thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread index does not exceed the array size
    if (i < n) {
        // Add the elements from array x and array y and store in y
        y[i] = x[i] + y[i];
    }
}