#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Optimize to use shared memory for better performance
    extern __shared__ float shared_feats[];

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < num_rows && j < num_cols) {
        // Load data into shared memory
        int shared_index = threadIdx.y * blockDim.x + threadIdx.x;
        shared_feats[shared_index] = feats[i * ldf + j];
        __syncthreads();  // Ensure all threads have written to shared memory

        // Compute square
        float f = shared_feats[shared_index];
        feats_sq[i * lds + j] = f * f;
    }
}