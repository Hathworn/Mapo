#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    const float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length / 2;

    // Use shared memory to improve memory access efficiency
    __shared__ float2 shared_data[CU1DBLOCK];
    
    for (int idx = thread_id; idx < half_length; idx += CU1DBLOCK) {
        if (idx == 0) continue;

        shared_data[threadIdx.x] = reinterpret_cast<const float2 *>(Ar)[idx];
        __syncthreads();
        
        float val_x = shared_data[threadIdx.x].x;
        float val_y = shared_data[threadIdx.x].y;
        
        float ret = val_x * val_x + val_y * val_y;
        if (use_power) {
            Aw[idx] = ret;
        } else {
            Aw[idx] = sqrtf(ret);
        }
        __syncthreads();
    }

    // Handle special case, only once per block
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabs(real);
            Aw[half_length] = fabs(im);
        }
    }
}