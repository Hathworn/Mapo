#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;

// Optimized: Coalesced memory access and offset checking
__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) { // Ensure index is within bounds
        c[index] = a[index] + b[index];
    }
}