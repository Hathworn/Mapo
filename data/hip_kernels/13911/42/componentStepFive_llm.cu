#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void componentStepFive(unsigned int *Q, unsigned int length, unsigned int *sprimtemp, unsigned int s) {
    unsigned int tid = blockIdx.y * gridDim.x * blockDim.x * blockDim.y
                     + blockIdx.x * blockDim.x * blockDim.y 
                     + threadIdx.y * blockDim.x 
                     + threadIdx.x;

    // Check if tid is within bounds once, and combine checks
    if (tid < length && Q[tid] == s) {
        // Use atomic operation to prevent race conditions
        atomicExch(sprimtemp, 1);
    }
}