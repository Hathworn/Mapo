#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constructCircuitGraphVertex(unsigned int * C, unsigned int * offset, unsigned int ecount, unsigned int * cv, unsigned int cvCount) {
    // Calculate global thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use vectorized access for speedup
    const unsigned int stride = blockDim.x * gridDim.x;
    for(unsigned int i = tid; i < ecount; i += stride){
        if(C[i] != 0){
            cv[offset[i]] = i;
        }
    }
}