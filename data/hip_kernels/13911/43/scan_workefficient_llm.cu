#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scan_workefficient(float *g_odata, float *g_idata, int n)
{
    // Dynamically allocated shared memory for scan kernels
    extern __shared__ float temp[];

    int thid = threadIdx.x;
    int offset = 1;

    // Load inputs into shared memory
    int ai = 2 * thid;
    int bi = 2 * thid + 1;
    temp[ai] = g_idata[ai];
    temp[bi] = g_idata[bi];

    // Build the sum in place up the tree
    for (int d = n >> 1; d > 0; d >>= 1)
    {
        __syncthreads();
        if (thid < d)
        {
            int addrA = offset * (2 * thid + 1) - 1;
            int addrB = offset * (2 * thid + 2) - 1;

            temp[addrB] += temp[addrA];
        }
        offset *= 2;
    }

    // Clear the last element
    if (thid == 0) temp[n - 1] = 0;

    // Traverse down the tree building the scan in place
    for (int d = 1; d < n; d *= 2)
    {
        offset >>= 1;
        __syncthreads();
        if (thid < d)
        {
            int addrA = offset * (2 * thid + 1) - 1;
            int addrB = offset * (2 * thid + 2) - 1;

            float t = temp[addrA];
            temp[addrA] = temp[addrB];
            temp[addrB] += t;
        }
    }

    __syncthreads();

    // Write results to global memory
    g_odata[ai] = temp[ai];
    g_odata[bi] = temp[bi];
}