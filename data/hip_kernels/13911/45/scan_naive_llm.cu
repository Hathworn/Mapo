#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan_naive(float *g_odata, float *g_idata, int n)
{
    // Dynamically allocated shared memory for scan kernels
    extern __shared__ float temp[];

    int thid = threadIdx.x;

    int pout = 0;
    int pin = 1;
    
    // Efficiently initialize shared memory
    temp[thid] = (thid > 0) ? g_idata[thid - 1] : 0;
    __syncthreads();

    // In-place scan using shared memory
    for (int offset = 1; offset < n; offset *= 2)
    {
        float val = temp[thid];
        if (thid >= offset)
            val += temp[thid - offset];
        __syncthreads();
        temp[thid] = val;
        __syncthreads();
    }

    // Write the result to global memory
    g_odata[thid] = temp[thid];
}