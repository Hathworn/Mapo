#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void markSegments(unsigned short* d_mark, unsigned int circuitGraphEdgeCount, unsigned int* d_cg_edge_start, unsigned int* d_cedgeCount, unsigned int circuitVertexSize) {
    // Calculate global thread index more efficiently
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int gridStride = blockDim.x * gridDim.x;

    // Use loop to handle larger data sizes
    while (tid < circuitVertexSize) {
        d_mark[d_cg_edge_start[tid]] = d_cedgeCount[tid];
        tid += gridStride;
    }
}