#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculateCircuitGraphVertexData(unsigned int *D, unsigned int *C, unsigned int ecount) {
    // Calculate global thread index
    unsigned int tid = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    
    if (tid < ecount) {
        unsigned int c = D[tid];
        // Use atomic operation to update C array
        atomicExch(&C[c], 1);
    }
}