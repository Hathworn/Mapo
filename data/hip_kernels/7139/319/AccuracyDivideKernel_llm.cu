#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    if (threadIdx.x == 0) { // Ensure division is done once using a single thread
        *accuracy /= N;
    }
}