#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define N 10

using namespace std;

__global__ void add(int *a, int *b, int *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread ID
    if (i < N)
        c[i] = a[i] + b[i];  // Perform addition if within bounds
}