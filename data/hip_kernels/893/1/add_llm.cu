#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;  // Calculate stride for grid stride loop

    for (int i = index; i < n; i += stride)  // Use grid stride loop for better performance
    {
        c[i] = a[i] + b[i];
    }
}