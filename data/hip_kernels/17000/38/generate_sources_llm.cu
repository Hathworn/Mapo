#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void generate_sources(hiprandState *state, int n, uint32_t *verts) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    if (idx < n) { // Check bounds before processing
        hiprandState local_state = state[idx];
        for (int id = idx; id < n; id += stride) {
            verts[id] = hiprand(&local_state);
        }
        state[idx] = local_state;
    }
}