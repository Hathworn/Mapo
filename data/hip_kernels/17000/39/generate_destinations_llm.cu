#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void generate_destinations(hiprandState *state, int n, const uint32_t *sources, uint32_t *destinations) {
    int first = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Initialize local state once outside loop for better performance
    hiprandState local_state = state[first];

    // Optimize the loop by calculating destination directly and reduce global memory access
    for (int id = first; id < n; id += stride) {
        // Use modulo operation directly on generated random value
        destinations[id] = sources[hiprand(&local_state) % n];
    }

    // Save the state back to global memory
    state[first] = local_state;
}