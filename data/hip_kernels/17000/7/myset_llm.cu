#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void myset(unsigned long long *p, unsigned long long v, long long n) {
    const long long tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Handle multiple assignments per thread based on stride
    for (long long i = tid; i < n; i += gridDim.x * blockDim.x) {
        p[i] = v;
    }
}