#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use each thread to divide a portion, assuming accuracy is an array
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        accuracy[idx] /= N;
    }
}