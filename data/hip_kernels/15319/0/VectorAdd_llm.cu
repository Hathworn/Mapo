#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define SIZE 1024

__global__ void VectorAdd(int *a, int *b, int *c, int n) {
    // Compute a unique index for each thread in the grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure that we do not read/write beyond the array bounds
    if (i < n) {
        // Perform vector addition for the valid index
        c[i] = a[i] + b[i];
        printf("i: %d, c[i] = %d, a[i] = %d, b[i] = %d\n", i, c[i], a[i], b[i]);
    }
}