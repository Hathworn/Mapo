#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use loop unrolling and shared memory to improve memory access efficiency
    __shared__ float s_outgrad[256];
    __shared__ float s_indata[256];

    if (tid < N) {
        s_outgrad[threadIdx.x] = outgrad[tid];
        s_indata[threadIdx.x] = indata[tid];

        __syncthreads();

        ingrad[tid] = s_indata[threadIdx.x] > 0 ? s_outgrad[threadIdx.x] : 0;
    }
}