#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_gradient_kernel( const int dim, const float* Y, const float* dY, float* dX) {
    // Adjust pointers for this block
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;

    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];
    
    float thread_sum = 0;

    // Perform a reduction to compute the inner product
    for (int i = idx; i < dim; i += blockDim.x) {
        thread_sum += dY[i] * Y[i];
    }
    
    // Store in shared buffer for reduction
    reduction_buffer[idx] = thread_sum;
    __syncthreads();

    // Perform parallel reduction using grid-reduction pattern
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (idx < stride) {
            reduction_buffer[idx] += reduction_buffer[idx + stride];
        }
        __syncthreads();
    }

    // Compute gradient using the reduction result
    float reduction_result = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - reduction_result);
    }
}