#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use an atomic operation to safely divide in parallel execution
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}