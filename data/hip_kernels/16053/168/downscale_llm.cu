#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii/d3;
    z = ii % d2;
    ii = ii/d2;
    y = ii % d1;
    ii = ii/d1;
    x = ii;
    w = w*scale_factor + off_x;
    z = z*scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x*d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii/d3;
    z = ii % d2;
    ii = ii/d2;
    y = ii % d1;
    ii = ii/d1;
    x = ii;
    w /= scale_factor;
    z /= scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x*d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x;
    ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
    if (ii >= no_elements) return;

    // Use registers for repetitive computations
    int sf = scale_factor * scale_factor; // Pre-calculate scale factor squared
    float gradInput_val = gradInput_data[ii]; // Cache current value for the current index

    int base_idx = translate_idx_inv(ii, d1, d2, d3, scale_factor, 0, 0); // Base index for offset calculations, more efficient

    for (int i = 0; i < sf; i++) { // Use direct computation instead of nested loops
        int ipidx = base_idx + (i % scale_factor) + (i / scale_factor) * d3; // Offset calculation simplified
        gradInput_val += gradOutput_data[ipidx]; // Accumulate result
    }

    gradInput_data[ii] = gradInput_val; // Store the accumulated result
}