#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;

    // Calculate write stride for coalesced access
    const int rowStride = tgtWidth * srcHeight;
    
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Calculate y and x using integer division and modulo
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        
        // Calculate source indices using modulo for tiling
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Write to target with coalesced accesses
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}