#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target,  const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    // Calculate global element index
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    shmem[threadIdx.x] = 0;

    // Loop with correct data boundary check
    if (eidx < numElements) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            shmem[threadIdx.x] += a[eidx] * b[eidx];
        }
    }

    __syncthreads();

    // Unrolling reduction without unnecessary barriers
    if (DP_BLOCKSIZE >= 512) {
        if (threadIdx.x < 256) shmem[threadIdx.x] += shmem[threadIdx.x + 256];
        __syncthreads();
    }
    if (DP_BLOCKSIZE >= 256) {
        if (threadIdx.x < 128) shmem[threadIdx.x] += shmem[threadIdx.x + 128];
        __syncthreads();
    }
    if (DP_BLOCKSIZE >= 128) {
        if (threadIdx.x < 64) shmem[threadIdx.x] += shmem[threadIdx.x + 64];
        __syncthreads();
    }

    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    // Write result to target
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}