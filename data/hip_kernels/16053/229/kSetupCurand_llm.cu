#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate unique thread index
    uint tidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Each thread gets a different seed with no offset
    hiprand_init(seed, tidx, 0, &state[tidx]);
}