#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void max_pool3d_forward(int B, int N, int M, int C, int K, const int* nnIndex, const int* nnCount, const float* input, float* output, int* maxIndex)
{
    int i = blockIdx.x;
    while(i < B) {
        int j = threadIdx.x;
        while(j < M * C) {
            int m = j / C;
            int c = j % C;
            int nnSize = nnCount[i * M + m];
            
            float maxVal = -FLT_MAX;
            int maxIdx = -1;

            for(int k = 0; k < nnSize; k++) {
                int n = nnIndex[i * M * K + m * K + k];
                float val = input[i * N * C + n * C + c];
                
                if (val > maxVal) {
                    maxVal = val;
                    maxIdx = n;
                }
            }

            output[i * M * C + j] = maxVal;
            maxIndex[i * M * C + j] = maxIdx;

            j += blockDim.x;
        }
        i += gridDim.x;
    }
}