#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_interpolate_backward(int B, int N, int M, int C, int K, const int* nnIndex, const int* nnCount, const float* gradOutput, float* gradInput)
{
    // Calculate global thread Id
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure thread is within bounds
    if (i < B && j < N * C)
    {
        int n = j / C;
        int c = j % C;
        int nnSize = nnCount[i * N + n];
        
        // Loop through neighbors
        for(int k = 0; k < nnSize; k++)
        {
            int m = nnIndex[i * N * K + n * K + k];
            // Atomic addition to avoid race conditions
            atomicAdd(&gradInput[i * M * C + m * C + c], gradOutput[i * N * C + j] / nnSize);
        }
    }
}