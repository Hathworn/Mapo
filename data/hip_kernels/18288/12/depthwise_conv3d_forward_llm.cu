#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void depthwise_conv3d_forward(int B, int N, int M, int C, int r, int K, const int* nnIndex, const int* nnCount, const int* binIndex, const float* input, const float* filter, float* output)
{
    // Launching threads to optimize memory access and parallel computation
    for(int i=blockIdx.x; i<B; i+=gridDim.x) 
    {
        // Use 2D block indexing for efficient memory access
        int tid = blockIdx.y * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.y;

        for(int j=tid; j<M*(C*r); j+=stride) 
        {
            int cout = j % (C*r);  // output channel ID
            int cin = cout / r;    // input channel ID
            int m = j / (C*r);     // output point ID
            int nnSize = nnCount[i*M + m];

            // Loop over neighbor points
            for(int k=0; k<nnSize; k++) 
            {
                int n = nnIndex[i*M*K + m*K + k];  // input point ID
                int f = binIndex[i*M*K + m*K + k];

                // Accumulate results for each output element
                output[i*M*C*r + j] += input[i*N*C + n*C + cin] * filter[f*C*r + cout] / nnSize;
            }
        }
    }
}