#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_interpolate_forward(int B, int N, int M, int C, int K, const int* nnIndex, const int* nnCount, const float* input, float* output)
{
    int i = blockIdx.x;
    int j = threadIdx.x;

    // Iterate over batches
    while(i < B) 
    {
        // Iterate over elements
        while(j < N * C) 
        {
            int n = j / C;
            int c = j % C;
            int nnSize = nnCount[i * N + n];
            float sum = 0.0f;

            // Accumulate neighbor contributions
            for(int k = 0; k < nnSize; k++)
            {
                int m = nnIndex[i * N * K + n * K + k];
                sum += input[i * M * C + m * C + c];
            }

            // Write the averaged value to output
            if (nnSize > 0)
            {
                output[i * N * C + j] = sum / nnSize;
            }
            
            j += blockDim.x; // Move to the next element
        }
        
        i += gridDim.x; // Move to the next batch
    }
}