#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void depthwise_input_backward(int B, int N, int M, int F, int C, int r, int K, 
                                         const int* nnIndex, const int* nnCount, const int* binIndex, 
                                         const float* input, const float* filter, const float* gradOutput, 
                                         float* gradInput) 
{
    int i = blockIdx.x; // Using blockIdx.x for indexing batch directly
    int j = blockIdx.y * blockDim.x + threadIdx.x; // Combined indexing within the block for 1D thread offset

    if (i < B && j < M * C * r) // Bounds check
    {
        int cout = j % (C * r); // output channel ID
        int cin = cout / r;     // input channel ID
        int m = j / (C * r);    // output point ID
        int nnSize = nnCount[i * M + m];

        for (int k = 0; k < nnSize; k++)
        {
            int n = nnIndex[i * M * K + m * K + k]; // input point ID
            int f = binIndex[i * M * K + m * K + k];

            // Optimize by precomputing common expressions
            float gradOutVal = gradOutput[i * M * C * r + j];
            float filterVal = filter[f * C * r + cout];
            float derIn = gradOutVal * filterVal / nnSize;

            atomicAdd(&gradInput[i * N * C + n * C + cin], derIn);
        }
    }
}