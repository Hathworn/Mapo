#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void avg_pool3d_forward(int B, int N, int M, int C, int K, const int* nnIndex, const int* nnCount, const float* input, float* output)
{
    // Utilize shared memory for nnCount to reduce global memory access
    extern __shared__ int shared_nnCount[];

    for (int i = blockIdx.x; i < B; i += gridDim.x)
    {
        // Load nnCount into shared memory
        if (threadIdx.x < M)
        {
            shared_nnCount[i * M + threadIdx.x] = nnCount[i * M + threadIdx.x];
        }
        __syncthreads();

        for (int j = threadIdx.x; j < M * C; j += blockDim.x)
        {
            int m = j / C;
            int c = j % C;
            int nnSize = shared_nnCount[i * M + m];

            // Use a local register to accumulate results before writing to global memory
            float sum = 0.0f;
            for (int k = 0; k < nnSize; k++)
            {
                int n = nnIndex[i * M * K + m * K + k];
                sum += input[i * N * C + n * C + c];
            }
            output[i * M * C + j] = sum / nnSize;
        }
    }
}