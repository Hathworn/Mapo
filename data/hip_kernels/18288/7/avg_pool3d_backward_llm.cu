#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void avg_pool3d_backward(int B, int N, int M, int C, int K, const int* nnIndex, const int* nnCount, const float* gradOutput, float* gradInput)
{
    int i = blockIdx.x;
    int j = threadIdx.x;
    
    // Ensure valid block and thread indexes
    if (i < B && j < M * C) {
        int m = j / C;
        int c = j % C;
        int nnSize = nnCount[i * M + m];
        
        // Check for non-zero neighbors
        if (nnSize > 0) {
            float scale = gradOutput[i * M * C + j] / nnSize;
            
            for (int k = 0; k < nnSize; k++) {
                int n = nnIndex[i * M * K + m * K + k]; // only neighbor, no bin indices, dimension=(B,M,K)
                atomicAdd(&gradInput[i * N * C + n * C + c], scale);
            }
        }
    }
}