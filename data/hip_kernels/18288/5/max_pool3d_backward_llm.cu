#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void max_pool3d_backward(int B, int N, int M, int C, const int* maxIndex, const float* gradOutput, float* gradInput)
{
    int i = blockIdx.x; // Use blockIdx.x for indexing over B
    int threadId = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Improved indexing with stride to ensure all elements are covered
    for (int j = threadId; j < M * C; j += stride)
    {
        int c = j % C;
        int n = maxIndex[i * M * C + j];
        atomicAdd(&gradInput[i * N * C + n * C + c], gradOutput[i * M * C + j]);
    }
}