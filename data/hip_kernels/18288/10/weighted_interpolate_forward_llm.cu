#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_interpolate_forward(int B, int N, int M, int C, int K, const int* nnIndex, const int* nnCount, const float* input, const float* weight, float* output)
{
    int i = blockIdx.x; // Process one batch per block
    int j = threadIdx.x + blockIdx.y * blockDim.x; // Linearize thread index for parallel processing

    if (i < B && j < N * C) // Ensure within bounds
    {
        int n = j / C;
        int c = j % C;
        int nnSize = nnCount[i * N + n];
        float sum = 0.0f; // Initialize accumulation variable

        for (int k = 0; k < nnSize; k++) // Accumulate weighted sums
        {
            int m = nnIndex[i * N * K + n * K + k];
            float w = weight[i * N * K + n * K + k];
            sum += input[i * M * C + m * C + c] * w;
        }

        output[i * N * C + j] = sum; // Store accumulated result
    }
}