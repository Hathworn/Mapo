#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_interpolate_backward(int B, int N, int M, int C, int K, const int* nnIndex, const int* nnCount, const float* gradOutput, const float* weight, float* gradInput)
{
    int i = blockIdx.x; // Each block processes one batch
    int n = threadIdx.x / C; // Determine n by thread index
    int c = threadIdx.x % C; // Compute c from thread index
    
    if (i < B && n < N) // Ensure within bounds
    {
        float gradOutVal = gradOutput[i * N * C + n * C + c]; // Precompute gradOutput
        int nnSize = nnCount[i * N + n];
        for (int k = 0; k < nnSize; k++)
        {
            int m = nnIndex[i * N * K + n * K + k];
            float w = weight[i * N * K + n * K + k];
            atomicAdd(&gradInput[i * M * C + m * C + c], gradOutVal * w); // Use precomputed gradOutput
        }
    }
}