#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void depthwise_filter_backward(int B, int N, int M, int F, int C, int r, int K, const int* nnIndex, const int* nnCount, const int* binIndex, const float* input, const float* gradOutput, float* gradFilter, int sharedMemSize, int startIdx)
{
    extern __shared__ float gradPerBlock[]; 
    // Initialize shared memory
    for (int i = threadIdx.x; i < sharedMemSize; i += blockDim.x) {
        gradPerBlock[i] = 0;
    }
    __syncthreads();
    
    // Calculate start indices
    int endIdx = sharedMemSize + startIdx;
    int globalThreadIdx = blockIdx.y * blockDim.x + threadIdx.x;
    
    // Loop through batches
    for (int i = blockIdx.x; i < B; i += gridDim.x) {
        // Iterate over elements with stride to cover all
        for (int j = globalThreadIdx; j < M * (C * r); j += blockDim.x * gridDim.y) {
            int cout = j % (C * r);
            int cin = cout / r;
            int m = j / (C * r);
            int nnSize = nnCount[i * M + m];
            
            // Loop through neighbors
            for (int k = 0; k < nnSize; k++) {
                int n = nnIndex[i * M * K + m * K + k];
                int f = binIndex[i * M * K + m * K + k];
                
                // Calculate gradient
                float derFilt = gradOutput[i * M * C * r + j] * input[i * N * C + n * C + cin] / nnSize;
                int currIdx = f * C * r + cout;
                
                // Use atomic operations for safe accumulation
                if ((currIdx >= startIdx) && (currIdx < endIdx)) {
                    atomicAdd(&gradPerBlock[currIdx - startIdx], derFilt);
                }
            }
        }
    }
    __syncthreads();
    
    // Accumulate results into global memory
    for (int i = threadIdx.x; i < sharedMemSize; i += blockDim.x) {
        atomicAdd(&gradFilter[i + startIdx], gradPerBlock[i]);
    }
}