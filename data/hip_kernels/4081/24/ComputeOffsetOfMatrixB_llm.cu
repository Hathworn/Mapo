#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    int32_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        // Calculate the offset and store it in the output array
        output[i] = -row_sum[blockIdx.x];
    }
}