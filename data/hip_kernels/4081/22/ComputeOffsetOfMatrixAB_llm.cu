#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
    // Using blockIdx.x only once outside the loop for efficiency
    int idx = blockIdx.x * N + threadIdx.x;
    int rowValue = row_sum[blockIdx.x];
    for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
        // Calculating index only once to reduce arithmetic operations
        int globalIdx = idx + i * blockDim.x;
        output[globalIdx] = K_A_B - rowValue - col_sum[i];
    }
}