#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Calculate global index to handle more threads than N
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Stride loop to cover all elements with given number of threads
    for (int32_t i = idx; i < N; i += blockDim.x * gridDim.x) {
        output[blockIdx.x * N + i] = -col_sum[i];
    }
}