#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    // Optimized to initialize state only for the current thread
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}