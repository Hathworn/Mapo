#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize memory access and thread mapping
__global__ void NormalizeOutput(const int num_elements, const int* __restrict__ original, int64_t* __restrict__ to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        // Use shared memory for frequent values to optimize access
        __shared__ int64_t s_batch_index, s_class_index;
        if (threadIdx.x == 0) {
            s_batch_index = batch_index;
            s_class_index = class_index;
        }
        __syncthreads();
        
        int64_t orig_value = static_cast<int64_t>(original[idx]);
        
        to_normalize[idx * 3] = s_batch_index;
        to_normalize[idx * 3 + 1] = s_class_index;
        to_normalize[idx * 3 + 2] = orig_value;
    }
}