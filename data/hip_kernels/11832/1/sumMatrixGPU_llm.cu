#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny) {
    // Calculate global thread ID for the 1D flattened matrix
    unsigned int idx = blockIdx.y * blockDim.y * nx + blockIdx.x * blockDim.x + threadIdx.y * nx + threadIdx.x;

    // Boundary check before operation
    if (idx < nx * ny) {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}