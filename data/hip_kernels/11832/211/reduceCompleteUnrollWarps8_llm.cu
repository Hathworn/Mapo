#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceCompleteUnrollWarps8(int *g_idata, int *g_odata, unsigned int n) {
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;
    
    // Unroll 8 with boundary check optimization
    int sum = 0;
    if (idx < n) sum = g_idata[idx];
    if (idx + blockDim.x < n) sum += g_idata[idx + blockDim.x];
    if (idx + 2 * blockDim.x < n) sum += g_idata[idx + 2 * blockDim.x];
    if (idx + 3 * blockDim.x < n) sum += g_idata[idx + 3 * blockDim.x];
    if (idx + 4 * blockDim.x < n) sum += g_idata[idx + 4 * blockDim.x];
    if (idx + 5 * blockDim.x < n) sum += g_idata[idx + 5 * blockDim.x];
    if (idx + 6 * blockDim.x < n) sum += g_idata[idx + 6 * blockDim.x];
    if (idx + 7 * blockDim.x < n) sum += g_idata[idx + 7 * blockDim.x];
    
    idata[tid] = sum;
    __syncthreads();

    // In-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();
    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();
    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();
    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();

    // Unrolling warp with volatile optimization
    if (tid < 32) {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}