#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Use shared memory to load data for faster access
    extern __shared__ float sharedMem[];
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        // Load data into shared memory
        sharedMem[threadIdx.y * blockDim.x + threadIdx.x] = MatA[idx];
        __syncthreads();  // Synchronize threads to ensure complete loading

        // Use shared memory for computation
        MatC[idx] = sharedMem[threadIdx.y * blockDim.x + threadIdx.x] + MatB[idx];
    }
}