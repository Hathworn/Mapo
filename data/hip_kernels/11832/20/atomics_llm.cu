#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomics(int *shared_var, int *values_read, int N, int iters)
{
    // Calculate the unique thread ID.
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if the thread ID is out of bounds.
    if (tid >= N) return;

    // Use atomicAdd to increment shared_var and store the original value in values_read.
    values_read[tid] = atomicAdd(shared_var, iters + 1);

    // Use atomicAdd directly within the same call to accumulate iters in shared_var.
    // Unrolling this operation as part of the single atomicAdd call above for optimization.
}