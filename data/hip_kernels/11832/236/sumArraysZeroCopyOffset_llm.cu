#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysZeroCopyOffset(float *A, float *B, float *C, const int N, int offset)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Calculate effective index once for reuse
    int effectiveIndex = i + offset; 
    
    // Check if effective index is within bounds
    if (effectiveIndex < N) {
        C[effectiveIndex] = A[effectiveIndex] + B[effectiveIndex];
    }
}