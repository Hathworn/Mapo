#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceSmemDyn(int *g_idata, int *g_odata, unsigned int n) 
{
    extern __shared__ int smem[];

    // Set thread ID
    unsigned int tid = threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Load data into shared memory with automatic bounds checking
    if (tid < n) smem[tid] = idata[tid];
    else smem[tid] = 0; // Initialize remaining smem to zero for proper reduction
    __syncthreads();

    // Reduce in shared memory with loop to eliminate repetitive code
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) 
    {
        if (tid < s) smem[tid] += smem[tid + s];
        __syncthreads();
    }

    // Unrolling last warp without extra volatile keyword
    if (tid < 32) 
    {
        smem[tid] += smem[tid + 32];
        smem[tid] += smem[tid + 16];
        smem[tid] += smem[tid + 8];
        smem[tid] += smem[tid + 4];
        smem[tid] += smem[tid + 2];
        smem[tid] += smem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}