#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeSmemUnrollPad(float *out, float *in, const int nx, const int ny)
{
    // Static 1D shared memory with padding
    __shared__ float tile[BDIMY * (BDIMX * 2 + IPAD)];

    // Coordinate in original matrix
    unsigned int ix = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Linear global memory index for original matrix
    unsigned int ti = iy * nx + ix;

    // Thread index in transposed block
    unsigned int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = bidx / blockDim.y;
    unsigned int icol = bidx % blockDim.y;

    // Coordinate in transposed matrix
    unsigned int ix2 = blockIdx.y * blockDim.y + icol;
    unsigned int iy2 = 2 * blockIdx.x * blockDim.x + irow;

    // Linear global memory index for transposed matrix
    unsigned int to = iy2 * ny + ix2;

    if (ix < nx && ix + blockDim.x < nx && iy < ny)
    {
        // Load two rows from global memory to shared memory
        unsigned int row_idx = threadIdx.y * (blockDim.x * 2 + IPAD) + threadIdx.x;
        tile[row_idx] = in[ti];
        tile[row_idx + BDIMX] = in[ti + BDIMX];

        // Thread synchronization
        __syncthreads();

        // Store two rows to global memory from two columns of shared memory
        unsigned int col_idx = icol * (blockDim.x * 2 + IPAD) + irow;
        out[to]             = tile[col_idx];
        out[to + ny * BDIMX] = tile[col_idx + BDIMX];
    }
}