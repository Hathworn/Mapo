#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Use shared memory for faster access
    __shared__ float sharedA[256];
    __shared__ float sharedB[256];

    if (i < n) {
        sharedA[threadIdx.x] = A[i];
        sharedB[threadIdx.x] = B[i];
    }

    __syncthreads();  // Synchronize threads to ensure data is loaded

    if (k < n) C[k] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
}