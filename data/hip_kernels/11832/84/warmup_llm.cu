#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float * __restrict__ A, float * __restrict__ B, float * __restrict__ C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Use __restrict__ keyword to enable better optimization.
    if (k < n) C[i] = A[k] + B[k];
}