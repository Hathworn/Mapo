#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceCompleteUnrollWarps8Sync(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    int sum = 0; // Accumulate directly to reduce memory access
    if (idx + 7 * blockDim.x < n)
    {
        sum += g_idata[idx];
        sum += g_idata[idx + blockDim.x];
        sum += g_idata[idx + 2 * blockDim.x];
        sum += g_idata[idx + 3 * blockDim.x];
        sum += g_idata[idx + 4 * blockDim.x];
        sum += g_idata[idx + 5 * blockDim.x];
        sum += g_idata[idx + 6 * blockDim.x];
        sum += g_idata[idx + 7 * blockDim.x];
        idata[tid] = sum;
    }
    else
    {
        idata[tid] = 0;
    }

    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();

    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();

    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();

    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();

    if (tid < 32) {
        volatile int *vsmem = idata; // Use volatile to avoid unnecessary sync
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}