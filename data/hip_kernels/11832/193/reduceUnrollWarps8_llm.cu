#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrollWarps8 (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // avoid shared memory bank conflicts using __restrict__ pointer
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    int sum = 0; // use a register to store partial sum
    if (idx + 7 * blockDim.x < n) {
        sum += g_idata[idx] + g_idata[idx + blockDim.x] + g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x];
        sum += g_idata[idx + 4 * blockDim.x] + g_idata[idx + 5 * blockDim.x] + g_idata[idx + 6 * blockDim.x] + g_idata[idx + 7 * blockDim.x];
    }
    
    __syncthreads();

    // in-place reduction using shared memory
    idata[tid] = sum;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // unrolling warp using volatile keyword for in-place reduction
    if (tid < 32) {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}