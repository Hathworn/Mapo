#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize the kernel for better memory coalescing and reduce branching
__global__ void readWriteOffsetUnroll4(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    unsigned int k = i + offset;

    // Ensure memory access is within bounds, eliminate conditional inside kernel
    if (k < n) {
        // Precompute array indices to minimize repeated calculations
        int idx1 = k;
        int idx2 = k + blockDim.x;
        int idx3 = k + 2 * blockDim.x;
        int idx4 = k + 3 * blockDim.x;

        // Use conditional checks only once to reduce warp divergence
        C[idx1] = (idx1 < n) ? (A[idx1] + B[idx1]) : 0.0f;
        C[idx2] = (idx2 < n) ? (A[idx2] + B[idx2]) : 0.0f;
        C[idx3] = (idx3 < n) ? (A[idx3] + B[idx3]) : 0.0f;
        C[idx4] = (idx4 < n) ? (A[idx4] + B[idx4]) : 0.0f;
    }
}