#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void host_api_kernel(float *randomValues, float *out, int N)
{
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;

    // Loop with stride equal to total number of threads
    for (int i = tid; i < N; i += nthreads)
    {
        // Load, process and store in out array
        out[i] = randomValues[i] * 2.0f;
    }
}