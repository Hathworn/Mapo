#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized 1D stencil kernel
__global__ void stencil_1d(float *in, float *out)
{
    // Shared memory with padding for halo
    __shared__ float smem[BDIM + 2 * RADIUS];

    // Calculate global and shared memory indices
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int sidx = threadIdx.x + RADIUS;

    // Load main block of data into shared memory
    smem[sidx] = in[idx];

    // Load halo regions
    if (threadIdx.x < RADIUS)
    {
        smem[sidx - RADIUS] = in[idx - RADIUS];
        smem[sidx + BDIM] = in[idx + BDIM];
    }

    // Synchronize threads
    __syncthreads();

    // Compute stencil
    float tmp = coef[0] * smem[sidx]; // Center coefficient
    #pragma unroll
    for (int i = 1; i <= RADIUS; i++)
    {
        tmp += coef[i] * (smem[sidx + i] - smem[sidx - i]);
    }

    // Store result
    out[idx] = tmp;
}