#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceInterleaved (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if(idx >= n) return;

    // in-place reduction in shared memory
    // Load data into shared memory for faster access
    extern __shared__ int sdata[];
    sdata[tid] = idata[tid];
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        // perform reduction in shared memory
        if (tid < stride)
        {
            sdata[tid] += sdata[tid + stride];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}