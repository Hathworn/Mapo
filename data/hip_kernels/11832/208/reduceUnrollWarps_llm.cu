#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrollWarps (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // unrolling 2
    if (idx + blockDim.x < n) g_idata[idx] += g_idata[idx + blockDim.x];
    
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x >> 1; stride > 32; stride >>= 1)
    {
        if (tid < stride)
        {
            // Simplified index
            idata[tid] += idata[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // unrolling last warp
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        // Use single loop for warp unrolling, reduces code repetition
        for (int step = 32; step >= 1; step >>= 1)
            vsmem[tid] += vsmem[tid + step];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}