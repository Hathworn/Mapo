#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d(float *in, float *out, int N) {
    // shared memory
    __shared__ float smem[BDIM + 2 * RADIUS];

    // index to global memory
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop until all elements are processed
    while (idx < N) {
        // index to shared memory for stencil calculation
        int sidx = threadIdx.x + RADIUS;

        // Read data from global memory into shared memory
        smem[sidx] = in[idx];

        // read halo part to shared memory with bounds checking
        if (threadIdx.x < RADIUS) {
            smem[sidx - RADIUS] = (idx >= RADIUS) ? in[idx - RADIUS] : 0.0f;  // Check bounds
            smem[sidx + BDIM] = (idx + BDIM < N) ? in[idx + BDIM] : 0.0f;    // Check bounds
        }

        // Synchronize (ensure all the data is available)
        __syncthreads();

        // Apply the stencil
        float tmp = 0.0f;

        #pragma unroll
        for (int i = 1; i <= RADIUS; i++) {
            tmp += coef[i] * (smem[sidx + i] - smem[sidx - i]);
        }

        // Store the result
        out[idx] = tmp;

        // Move to the next part of the input array
        idx += gridDim.x * blockDim.x;
    }
}