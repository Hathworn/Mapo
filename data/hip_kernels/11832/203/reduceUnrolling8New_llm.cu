#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling8New (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    
    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;
    
    // unrolling 8 with boundary check
    int tmp = 0;
    if (idx + 7 * blockDim.x < n)
    {
        #pragma unroll
        for (int i = 0; i < 8; i++)
        {
            tmp += g_idata[idx + i * blockDim.x];
        }
    }
    else
    {
        for (int i = 0; i < 8 && idx + i * blockDim.x < n; i++)
        {
            tmp += g_idata[idx + i * blockDim.x];
        }
    }

    // Write the result of the unrolled addition to the block's local memory
    idata[tid] = tmp;

    __syncthreads();

    // in-place reduction in shared memory
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        
        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0) 
        g_odata[blockIdx.x] = idata[0];
}