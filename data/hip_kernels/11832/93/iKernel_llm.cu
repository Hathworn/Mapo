#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iKernel(float *src, float *dst)
{
    // Calculate the global thread index
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Optimize memory access by checking idx boundary
    if (idx < gridDim.x * blockDim.x) {
        dst[idx] = src[idx] * 2.0f;
    }
}