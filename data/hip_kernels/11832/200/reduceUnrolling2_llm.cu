#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling2 (int *g_idata, int *g_odata, unsigned int n) {
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    
    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // Unrolling 2 with boundary check
    if (idx + blockDim.x < n) {
        g_idata[idx] += g_idata[idx + blockDim.x];
    }
    
    __syncthreads();

    // In-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); // Synchronize within threadblock after each reduction step
    }

    // Write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}