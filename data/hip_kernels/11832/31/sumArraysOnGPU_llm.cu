#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop to handle more elements with fewer kernel launches
    for (; i < N; i += blockDim.x * gridDim.x) {
        C[i] = A[i] + B[i];
    }
}