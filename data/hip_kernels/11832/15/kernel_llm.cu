#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *F, double *D)
{
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single thread to initialize shared data for efficiency
    if (tid == 0)
    {
        *F = 12.1f;  // Use 'f' suffix for float literals
        *D = 12.1;   // Double initialization is fine as-is
    }
}