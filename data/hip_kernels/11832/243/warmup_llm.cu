#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *A, float *B, float *C, const int N)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for better performance
    __shared__ float sA[TILE_SIZE];
    __shared__ float sB[TILE_SIZE];

    // Load data into shared memory
    if (i < N)
    {
        sA[threadIdx.x] = A[i];
        sB[threadIdx.x] = B[i];
    }
    __syncthreads();

    // Ensure boundary check and perform addition
    if (i < N)
    {
        C[i] = sA[threadIdx.x] + sB[threadIdx.x];
    }
}
```
