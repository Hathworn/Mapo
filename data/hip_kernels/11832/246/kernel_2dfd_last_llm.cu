#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_2dfd_last(float *g_u1, float *g_u2, const int nx, const int iStart, const int iEnd)
{
    // global to slice : global grid idx for (x,y) plane
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;

    // smem idx for current point
    unsigned int stx = threadIdx.x + NPAD;
    unsigned int idx = ix + iStart * nx;

    // shared memory for u2 with size [4+16+4][4+16+4]
    __shared__ float tile[BDIMX + NPAD2];

    const float alpha = 0.12f;

    // register for y value
    float yval[9];

    // Precompute yvals
    #pragma unroll
    for (int i = 0; i < 8; i++) yval[i] = g_u2[idx + (i - 4) * nx];

    // to be used in z loop
    int iskip = NPAD * nx;

    #pragma unroll 9
    for (int iy = iStart; iy < iEnd; iy++)
    {
        // load front3 into yval[8]
        yval[8] = g_u2[idx + iskip];

        // Load shared memory
        if (threadIdx.x < NPAD) {
            tile[threadIdx.x] = g_u2[idx - NPAD];
            tile[stx + BDIMX] = g_u2[idx + BDIMX];
        }

        // Load the center value into shared memory
        tile[stx] = yval[4];
        __syncthreads();

        // Compute only within the valid range
        if ((ix >= NPAD) && (ix < nx - NPAD))
        {
            // Apply the finite difference
            float tmp = coef[0] * tile[stx] * 2.0f;

            #pragma unroll // Unroll loop for x dimensions
            for (int d = 1; d <= 4; d++) {
                tmp += coef[d] * (tile[stx - d] + tile[stx + d]);
            }

            #pragma unroll // Unroll loop for y dimensions
            for (int d = 1; d <= 4; d++) {
                tmp += coef[d] * (yval[4 - d] + yval[4 + d]);
            }

            // Time stepping
            g_u1[idx] = yval[4] + yval[4] - g_u1[idx] + alpha * tmp;
        }

        // Shift y values up
        #pragma unroll 8
        for (int i = 0; i < 8; i++) {
            yval[i] = yval[i + 1];
        }

        // Advance global index
        idx += nx;
        __syncthreads();
    }
}
```
