#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Row(float *out, float *in, const int nx, const int ny)
{
    // Calculate indexes with less operations
    unsigned int ix = (blockDim.x * blockIdx.x + threadIdx.x) * 4;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    
    if (ix < nx && iy < ny)
    {
        unsigned int ti = iy * nx + ix; // access in rows
        unsigned int to = ix * ny + iy; // access in columns
        
        // Reuse memory locations to avoid recalculating offsets
        out[to] = in[ti];
        
        ti += blockDim.x;
        to += ny * blockDim.x;
        out[to] = in[ti];
        
        ti += blockDim.x;
        to += ny * blockDim.x;
        out[to] = in[ti];
        
        ti += blockDim.x;
        to += ny * blockDim.x;
        out[to] = in[ti];
    }
}