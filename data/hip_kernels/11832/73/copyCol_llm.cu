#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate 1D block and grid indices using threadIdx and blockIdx for coalesced memory access
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = gridDim.x * blockDim.x;

    // Loop with a stride to cover the entire array
    while (idx < nx * ny)
    {
        out[idx] = in[idx];
        idx += stride;
    }
}