#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmem(int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // boundary check
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= n) return;

    // load data to shared memory
    int *idata = g_idata + blockIdx.x * blockDim.x;
    smem[tid] = idata[tid];  
    __syncthreads();

    // in-place reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) smem[tid] += smem[tid + s];
        __syncthreads();
    }

    // unrolling warp
    if (tid < 32) {
        volatile int* vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}