#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrollWarps8 (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // Unrolling 8: check boundary once and perform reduction
    if (idx + 7 * blockDim.x < n)
    {
        int sum = g_idata[idx] + g_idata[idx + blockDim.x] +
                  g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x] +
                  g_idata[idx + 4 * blockDim.x] + g_idata[idx + 5 * blockDim.x] +
                  g_idata[idx + 6 * blockDim.x] + g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = sum;
    }

    __syncthreads();

    // In-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();  // synchronize within threadblock
    }

    // Unrolling final 32 threads
    if (tid < 32)
    {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}