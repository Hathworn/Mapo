#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate global thread index
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Check if thread is within bounds
    if (ix >= nx || iy >= ny) return;
    
    // Calculate linear index
    unsigned int idx = iy * nx + ix;
    
    // Perform computation
    MatC[idx] = MatA[idx] + MatB[idx];
}