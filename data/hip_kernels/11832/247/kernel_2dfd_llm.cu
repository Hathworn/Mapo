#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_2dfd(float *g_u1, float *g_u2, const int nx, const int iStart, const int iEnd)
{
    // Calculate global x index
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate shared memory index
    unsigned int stx = threadIdx.x + NPAD;
    unsigned int idx = ix + iStart * nx;

    // Shared memory for the x dimension
    __shared__ float line[BDIMX + NPAD2];

    // Coefficient related to physical properties
    const float alpha = 0.12f;

    // Register for y values
    float yval[9];

    // Initialize yval with surrounding values
    #pragma unroll
    for (int i = 0; i < 8; i++) yval[i] = g_u2[idx + (i - 4) * nx];

    int iskip = NPAD * nx;

    #pragma unroll 9
    for (int iy = iStart; iy < iEnd; iy++)
    {
        // Fetch next y value
        yval[8] = g_u2[idx + iskip];

        // Load halo parts into shared memory
        if (threadIdx.x < NPAD)
        {
            line[threadIdx.x] = g_u2[idx - NPAD];
            line[stx + BDIMX] = g_u2[idx + BDIMX];
        }

        line[stx] = yval[4];
        __syncthreads();

        // Apply the finite difference stencil
        if ((ix >= NPAD) && (ix < nx - NPAD))
        {
            // Center point calculated first
            float tmp = coef[0] * line[stx] * 2.0f;

            // Unroll loop for neighboring points
            #pragma unroll
            for (int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (line[stx - d] + line[stx + d]);
            }

            #pragma unroll
            for (int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (yval[4 - d] + yval[4 + d]);
            }

            // Update value with time dimension
            g_u1[idx] = yval[4] + yval[4] - g_u1[idx] + alpha * tmp;
        }

        // Shift yval buffer for next iteration
        #pragma unroll 8
        for (int i = 0; i < 8 ; i++)
        {
            yval[i] = yval[i + 1];
        }

        // Advance to the next global index
        idx += nx;
        __syncthreads();
    }
}