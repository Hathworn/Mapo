#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysZeroCopy(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Unrolling to improve throughput
    int gridSize = blockDim.x * gridDim.x;
    while (i < N) {
        C[i] = A[i] + B[i];
        i += gridSize;
    }
}