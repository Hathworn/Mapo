#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceGmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 4;

    // unrolling 4
    int sum = 0;
    if (idx + 3 * blockDim.x < n)
    {
        sum += g_idata[idx];
        sum += g_idata[idx + blockDim.x];
        sum += g_idata[idx + 2 * blockDim.x];
        sum += g_idata[idx + 3 * blockDim.x];
    }
    idata[tid] = sum;
    
    __syncthreads();

    // in-place reduction in global memory
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            idata[tid] += idata[tid + s];
        }
        __syncthreads();
    }

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = idata; // use volatile to prevent optimization issues
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}