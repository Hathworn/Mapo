#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaiveRow(float *out, float *in, const int nx, const int ny)
{
    // Calculate transposed index using cooperative thread array dimension
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Check within bounds and perform transposition
    if (ix < nx && iy < ny)
    {
        // Optimize by reducing global memory accesses
        unsigned int inputIndex = iy * nx + ix;
        unsigned int outputIndex = ix * ny + iy;
        out[outputIndex] = in[inputIndex];
    }
}