#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate 2D thread index
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Linearize 2D grid to 1D index
    if (ix < nx && iy < ny) {
        unsigned int idx = iy * nx + ix; // Calculation inside the if condition
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}