#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemUnrollDyn(int *g_idata, int *g_odata, unsigned int n)
{
    extern __shared__ int smem[];

    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // Unrolling 4 and boundary check combined 
    int tmpSum = 0;
    if (idx < n) tmpSum += g_idata[idx];
    if (idx + blockDim.x < n) tmpSum += g_idata[idx + blockDim.x];
    if (idx + 2 * blockDim.x < n) tmpSum += g_idata[idx + 2 * blockDim.x];
    if (idx + 3 * blockDim.x < n) tmpSum += g_idata[idx + 3 * blockDim.x];

    smem[tid] = tmpSum;
    __syncthreads();

    // In-place reduction using loop to reduce unroll duplication 
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            smem[tid] += smem[tid + s];
        }
        __syncthreads();
    }

    // Unrolling warp
    if (tid < 32) {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}