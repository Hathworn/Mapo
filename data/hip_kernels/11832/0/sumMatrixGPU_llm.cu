#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Use shared memory to optimize memory access patterns
    __shared__ float shared_A[BLOCK_DIM_Y][BLOCK_DIM_X];
    __shared__ float shared_B[BLOCK_DIM_Y][BLOCK_DIM_X];

    // Calculate local thread indices
    unsigned int local_ix = threadIdx.x;
    unsigned int local_iy = threadIdx.y;

    // Calculate global indices
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    // Load data into shared memory if within bounds
    if (ix < nx && iy < ny) {
        shared_A[local_iy][local_ix] = MatA[idx];
        shared_B[local_iy][local_ix] = MatB[idx];
    }
    __syncthreads(); // Ensure all threads have loaded their data

    // Calculate the result using shared memory
    if (ix < nx && iy < ny) {
        MatC[idx] = shared_A[local_iy][local_ix] + shared_B[local_iy][local_ix];
    }
}