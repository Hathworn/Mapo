#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeOffsetUnroll2(float *A, float *B, float *C, const int n, int offset)
{
    // Compute the global thread index
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int k = i + offset;

    // Use unrolled loop with shared memory to increase the memory transaction efficiency
    if (k < n)
    {
        C[k] = A[k] + B[k];
        
        // Unroll the loop to handle two elements per thread
        if (k + blockDim.x < n)
        {
            C[k + blockDim.x] = A[k + blockDim.x] + B[k + blockDim.x];
        }
    }
}