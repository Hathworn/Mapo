#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(int *MatA, int *MatB, int *MatC, int nx, int ny)
{
    // Calculate global thread coordinates
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Exit if thread is outside matrix bounds
    if (ix >= nx || iy >= ny) return;

    // Compute flattened index
    unsigned int idx = iy * nx + ix;

    // Sum corresponding elements of MatA and MatB
    MatC[idx] = MatA[idx] + MatB[idx];
}