#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceInterleaved (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if(idx >= n) return;

    // In-place reduction in shared memory for faster access
    __shared__ int shared_data[1024]; // Assuming blockDim.x <= 1024, adjust if necessary
    shared_data[tid] = (idx < n) ? idata[tid] : 0;
    __syncthreads();

    // Unroll the loop for better parallel reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = shared_data[0];
}