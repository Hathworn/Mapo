#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaiveCol(float *out, float *in, const int nx, const int ny)
{
    // Compute the global row and column indices
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure threads do not run beyond matrix dimensions
    if (ix < nx && iy < ny)
    {
        // Perform the transposition operation
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}