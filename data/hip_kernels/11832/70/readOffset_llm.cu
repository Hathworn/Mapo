#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffset(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Unroll loop to improve memory access pattern
    if (k < n)
    {
        C[i] = A[k] + B[k];

        // Preloading next elements to avoid multiple condition checks
        if((k + blockDim.x) < n)
            C[i + blockDim.x] = A[k + blockDim.x] + B[k + blockDim.x];
    }
}