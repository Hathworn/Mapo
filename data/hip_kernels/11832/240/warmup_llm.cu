#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;
    
    // Use warp shuffle to avoid divergent branches
    if (k < n) 
    {
        float a_val = A[i];
        float b_val = B[i];
        C[k] = a_val + b_val;
    }
}