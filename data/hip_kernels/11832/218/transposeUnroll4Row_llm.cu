#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeUnroll4Row(float *out, float *in, const int nx, const int ny)
{
    // Calculate global row and column indices
    unsigned int ix = blockIdx.x * (blockDim.x * 4) + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        unsigned int ti = iy * nx + ix; // Compute input index for row-major access
        unsigned int to = ix * ny + iy; // Compute output index for column-major access

        // Copy 4 elements in a coalesced manner
        out[to] = in[ti];
        out[to + ny * blockDim.x] = in[ti + blockDim.x];
        out[to + ny * 2 * blockDim.x] = in[ti + 2 * blockDim.x];
        out[to + ny * 3 * blockDim.x] = in[ti + 3 * blockDim.x];
    }
}