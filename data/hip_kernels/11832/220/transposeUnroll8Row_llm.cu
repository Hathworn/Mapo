#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeUnroll8Row(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 8 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Calculate initial indices
    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    // Boundary check to ensure threads don't access out of bounds memory
    if (ix + 7 * blockDim.x < nx && iy < ny)
    {
        // Prefetch data to reduce repetitive global memory access
        float input[8];
        #pragma unroll
        for (int i = 0; i < 8; ++i) {
            input[i] = in[ti + i * blockDim.x];
        }

        // Transfer the prefetched data to the output
        #pragma unroll
        for (int i = 0; i < 8; ++i) {
            out[to + i * ny * blockDim.x] = input[i];
        }
    }
}