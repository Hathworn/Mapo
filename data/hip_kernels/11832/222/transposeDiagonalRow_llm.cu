#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeDiagonalRow(float *out, float *in, const int nx, const int ny)
{
    // Calculate the unique block index using blockIdx.y to improve memory access pattern
    unsigned int blk_x = blockIdx.x;
    unsigned int blk_y = blockIdx.y;

    // Compute the global thread coordinates
    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    // Check if within bounds before accessing memory
    if (ix < nx && iy < ny)
    {
        // Perform the diagonal transpose efficiently
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}