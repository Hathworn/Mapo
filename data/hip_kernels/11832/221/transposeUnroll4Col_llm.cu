#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeUnroll4Col(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Accessing input and output with coalesced memory access pattern
    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        const unsigned int ti = iy * nx + ix;
        const unsigned int to = ix * ny + iy;
        
        float temp0 = in[to];
        float temp1 = in[to + blockDim.x * ny];
        float temp2 = in[to + 2 * blockDim.x * ny];
        float temp3 = in[to + 3 * blockDim.x * ny];

        out[ti]                = temp0;
        out[ti +   blockDim.x] = temp1;
        out[ti + 2 * blockDim.x] = temp2;
        out[ti + 3 * blockDim.x] = temp3;
    }
}