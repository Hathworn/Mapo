#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void iKernel(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use if condition outside memory access to reduce warp divergence
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}