#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_add_wavelet(float *g_u2, float wavelets, const int nx, const int ny, const int ngpus)
{
    // Calculate global grid index for (x,y) plane
    int ipos = (ngpus == 2 ? ny - 10 : ny / 2 - 10);
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Combine calculation of idx and condition check to reduce redundant operations
    if (ix == nx / 2) {
        g_u2[ipos * nx + ix] += wavelets;
    }
}