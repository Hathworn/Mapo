#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyRow(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Using shared memory to optimize memory accesses
    __shared__ float tile[BLOCK_SIZE_Y][BLOCK_SIZE_X];

    if (ix < nx && iy < ny)
    {
        tile[threadIdx.y][threadIdx.x] = in[iy * nx + ix];
        __syncthreads(); // Ensure all threads have copied their data

        out[iy * nx + ix] = tile[threadIdx.y][threadIdx.x];
    }
}