#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighboredLess (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + tid;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if(idx >= n) return;

    // In-place reduction in shared memory for faster access
    extern __shared__ int sdata[];
    sdata[tid] = idata[tid];
    __syncthreads();

    // Perform reduction
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads(); // Ensure all threads have updated their values
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}