#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void refill_randoms(float *dRand, int N, hiprandState *states)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;
    hiprandState localState = states[tid];  // Use a local copy of the state

    for (int i = tid; i < N; i += nthreads)
    {
        dRand[i] = hiprand_uniform(&localState);  // Use local state
    }

    states[tid] = localState;  // Save the updated state back
}