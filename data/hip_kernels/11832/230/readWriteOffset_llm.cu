#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readWriteOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global index incorporating offset
    unsigned int k = blockIdx.x * blockDim.x + threadIdx.x + offset;

    // Check boundary within 'if' to avoid unnecessary calculations
    if (k < n) 
    {
        C[k] = A[k] + B[k];
    }
}