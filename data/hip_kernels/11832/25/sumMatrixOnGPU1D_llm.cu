#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate global thread index
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Use a stride loop to ensure all elements are processed
    int totalSize = nx * ny;
    while (idx < totalSize) {
        MatC[idx] = MatA[idx] + MatB[idx];
        idx += blockDim.x * gridDim.x; // Move to the next set of elements
    }
}