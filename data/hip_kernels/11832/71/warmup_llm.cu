#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(float *out, float *in, const int nx, const int ny)
{
    // Calculate unique index using grid stride loop
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Combine index to directly access the array element
    unsigned int idx = iy * nx + ix;

    // Access and assign when indices are within bounds, using grid stride loop
    for (unsigned int i = idx; i < nx * ny; i += blockDim.x * gridDim.x)
    {
        out[i] = in[i];
    }
}