#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate row index
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    // Calculate column index
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int idx = iy * nx + ix;
    
    // Check boundaries and perform addition
    if (ix < nx && iy < ny) {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}