#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_reduction(int *shared_var, int *input_values, int N, int iters)
{
    __shared__ int local_mem[256]; // Shared memory for local block reduction
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;
    
    // Load data into shared memory
    if (tid < N) {
        local_mem[local_tid] = input_values[tid];
    }
    
    // Ensure all threads have written to shared memory before reduction
    __syncthreads();
    
    // Perform reduction only if needed
    for (int iter = 0; iter < iters; iter++) {
        for (int s = (blockDim.x >> 1); s > 0; s >>= 1) {
            if (local_tid < s && tid + s < N) {
                local_mem[local_tid] += local_mem[local_tid + s];
            }
            __syncthreads(); // Synchronize within thread block
        }
        
        // Use atomic add to accumulate block sum to global variable
        if (local_tid == 0) {
            atomicAdd(shared_var, local_mem[0]);
        }
        
        // Ensure all threads are done with atomic operation before next iteration
        __syncthreads();
    }
}