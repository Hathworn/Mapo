#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeSmemUnroll(float *out, float *in, const int nx, const int ny)
{
    // Reuse shared memory for 2D tile
    __shared__ float tile[BDIMY * BDIMX * 2];

    // Original matrix coordinates
    unsigned int ix = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Linear index in original matrix
    unsigned int ti = iy * nx + ix;

    // Transposed block thread index
    unsigned int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = bidx / blockDim.y;
    unsigned int icol = bidx % blockDim.y;

    // Transposed matrix coordinates
    unsigned int ix2 = blockIdx.y * blockDim.y + icol;
    unsigned int iy2 = 2 * blockIdx.x * blockDim.x + irow;

    // Linear index in transposed matrix
    unsigned int to = iy2 * ny + ix2;

    // Boundary check for loading
    if (ix + blockDim.x < nx && iy < ny)
    {
        // Coalesced global to shared memory load
        unsigned int row_idx = 2 * threadIdx.y * blockDim.x + threadIdx.x;
        tile[row_idx]       = in[ti];
        tile[row_idx + BDIMX] = in[ti + BDIMX];

        // Synchronization for shared memory
        __syncthreads();

        // Shared to global memory store
        unsigned int col_idx = icol * blockDim.x * 2 + irow;
        if (ix2 < ny && iy2 < nx)  // Boundary check for storing
        {
            out[to] = tile[col_idx];
            out[to + ny * BDIMX] = tile[col_idx + BDIMX];
        }
    }
}