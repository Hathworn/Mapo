#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysZeroCopyWithUVAOffset(float *A, float *B, float *C, const int N, int offset)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = i + offset;  // Pre-compute index with offset

    if (idx < N + offset)  // Adjust boundary condition
    {
        C[idx] = A[idx] + B[idx];  // Use pre-computed index
    }
}