#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeRow(float *out, float *in, const int nx, const int ny) 
{
    // Optimized: Flattened 2D grid and block indices into a linear index.
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny) {
        // Optimized: Calculate the linear index directly.
        unsigned int input_index = iy * nx + ix;
        unsigned int output_index = ix * ny + iy;
        out[output_index] = in[input_index];
    }
}