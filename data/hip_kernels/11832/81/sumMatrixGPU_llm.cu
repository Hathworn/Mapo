#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate flattened index for the current thread
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;

    // Avoid out-of-bounds memory access
    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}