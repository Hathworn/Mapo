#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BDIM 256
#define RADIUS 3
__constant__ float coef[RADIUS+1];

__global__ void stencil_1d(float *in, float *out)
{
    // shared memory
    __shared__ float smem[BDIM + 2 * RADIUS];

    // index to global memory
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // index to shared memory for stencil calculation
    int sidx = threadIdx.x + RADIUS;

    // Read data from global memory into shared memory
    smem[sidx] = in[idx];

    // read halo part to shared memory
    if (threadIdx.x < RADIUS)
    {
        smem[sidx - RADIUS] = in[max(idx - RADIUS, 0)];    // Boundary check to avoid access out of bounds
        smem[sidx + BDIM] = in[min(idx + BDIM, gridDim.x * blockDim.x - 1)];  // Boundary check for upper limit
    }

    // Synchronize (ensure all the data is available)
    __syncthreads();

    // Apply the stencil
    float tmp = 0.0f;

    #pragma unroll
    for (int i = 1; i <= RADIUS; i++)
    {
        tmp += coef[i] * (smem[sidx + i] - smem[sidx - i]);
    }

    // Store the result
    out[idx] = tmp;
}