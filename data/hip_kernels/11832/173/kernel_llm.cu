#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *g_data, float value)
{
    // Use shared memory for better data locality
    __shared__ float shared_data[256]; // Assumes blockDim.x is 256 or less

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int localIdx = threadIdx.x;

    // Load data into shared memory
    shared_data[localIdx] = g_data[idx];
    __syncthreads(); // Ensure all loads are completed

    // Perform computation using shared memory
    shared_data[localIdx] += value;

    __syncthreads(); // Ensure all computations are done

    // Store the result back to global memory
    g_data[idx] = shared_data[localIdx];
}