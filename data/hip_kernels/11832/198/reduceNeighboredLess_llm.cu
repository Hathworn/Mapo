#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighboredLess (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if(idx >= n) return;

    // In-place reduction in shared memory
    __shared__ int sdata[1024]; // Assumes blockDim.x <= 1024
    sdata[tid] = idata[tid];
    __syncthreads();

    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int index = 2 * stride * tid;
        if (index < blockDim.x) {
            sdata[index] += sdata[index + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}