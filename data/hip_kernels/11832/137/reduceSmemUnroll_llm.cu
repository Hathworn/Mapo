#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // static shared memory
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // global index, 4 blocks of input data processed at a time
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // unrolling 4 blocks
    int tmpSum = 0;

    // boundary check
    if (idx < n)
    {
        tmpSum = g_idata[idx];
        if (idx + blockDim.x < n) tmpSum += g_idata[idx + blockDim.x];
        if (idx + 2 * blockDim.x < n) tmpSum += g_idata[idx + 2 * blockDim.x];
        if (idx + 3 * blockDim.x < n) tmpSum += g_idata[idx + 3 * blockDim.x];
    }

    smem[tid] = tmpSum;
    __syncthreads();

    // in-place reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
    {
        if (tid < s) smem[tid] += smem[tid + s];
        __syncthreads();
    }

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}