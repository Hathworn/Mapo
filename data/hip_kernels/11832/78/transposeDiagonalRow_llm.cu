#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalRow(float *out, float *in, const int nx, const int ny)
{
    // Calculate transposed block index
    unsigned int blk_y = blockIdx.y;
    unsigned int blk_x = blockIdx.x;

    // Calculate the global thread coordinates
    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    // Perform the transpose operation when within bounds
    if (ix < nx && iy < ny)
    {
        out[iy + ix * ny] = in[ix + iy * nx];
    }
}