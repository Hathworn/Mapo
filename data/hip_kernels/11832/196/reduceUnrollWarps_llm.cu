#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrollWarps(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // Load data into shared memory
    __shared__ int sdata[1024]; // Assume blockDim.x <= 1024
    if (idx < n) {
        sdata[tid] = g_idata[idx] + (idx + blockDim.x < n ? g_idata[idx + blockDim.x] : 0);
    } else {
        sdata[tid] = 0;
    }
    __syncthreads();

    // In-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1)
    {
        if (tid < stride)
        {
            sdata[tid] += sdata[tid + stride];
        }
        // Synchronize within threadblock
        __syncthreads();
    }

    // Unrolling last warp
    if (tid < 32)
    {
        volatile int *vsmem = sdata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}