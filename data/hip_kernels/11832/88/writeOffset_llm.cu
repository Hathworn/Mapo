#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeOffset(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Ensure k is within bounds
    if (k < n) {
        // Use shared memory for better memory access patterns
        extern __shared__ float sharedA[];
        extern __shared__ float sharedB[];

        // Load data into shared memory
        sharedA[threadIdx.x] = A[i];
        sharedB[threadIdx.x] = B[i];
        __syncthreads();

        // Perform addition using shared memory
        C[k] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
    }
}