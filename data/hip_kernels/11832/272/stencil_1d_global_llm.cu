#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencil_1d_global (float* in, float* out, float * dcoef)
{
    // shared memory
    __shared__ float smem[BDIM + 2 * RADIUS];

    // index to global memory
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // index to shared memory for stencil calculation
    int sidx = threadIdx.x + RADIUS;

    // Read data from global memory into shared memory
    if (idx < N) {
        smem[sidx] = in[idx];
        
        // read halo part to shared memory if within bounds
        if (threadIdx.x < RADIUS) {
            smem[sidx - RADIUS] = (idx >= RADIUS) ? in[idx - RADIUS] : 0.0f;  // Prevent out-of-bounds access
            smem[sidx + BDIM] = (idx + BDIM < N) ? in[idx + BDIM] : 0.0f;      // Prevent out-of-bounds access
        }
    }

    // Synchronize (ensure all the data is available)
    __syncthreads();

    // Apply the stencil only if idx is within the valid range
    if (idx < N) {
        float tmp = 0.0f;
        #pragma unroll
        for (int i = 1; i <= RADIUS; i++) {
            tmp += dcoef[i] * (smem[sidx + i] - smem[sidx - i]);
        }

        // Store the result
        out[idx] = tmp;
    }
}