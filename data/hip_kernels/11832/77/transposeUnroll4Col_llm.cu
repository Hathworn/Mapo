#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Col(float *out, float *in, const int nx, const int ny)
{
    // Calculate global thread coordinates
    unsigned int ix = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate matrix indices
    unsigned int ti = iy * nx + ix;
    unsigned int to = ix * ny + iy;

    // Optimization: Check boundaries once to reduce divergence
    if (ix < nx - 3 * blockDim.x && iy < ny)
    {
        // Use registers to reduce global memory access
        float4 data;
        data.x = in[to];
        data.y = in[to + blockDim.x * ny];
        data.z = in[to + 2 * blockDim.x * ny];
        data.w = in[to + 3 * blockDim.x * ny];

        // Write to output in burst mode
        out[ti] = data.x;
        out[ti + blockDim.x] = data.y;
        out[ti + 2 * blockDim.x] = data.z;
        out[ti + 3 * blockDim.x] = data.w;
    }
}