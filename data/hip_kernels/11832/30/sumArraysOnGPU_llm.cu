#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Utilize blockIdx for larger arrays

    if (i < N) C[i] = A[i] + B[i]; // Process elements within the valid range
}