#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeDiagonalCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate transposed block coordinates
    unsigned int blk_x = blockIdx.x;
    unsigned int blk_y = blockIdx.y;

    // Calculate transposed global thread coordinates
    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    // Ensure indices are within bounds
    if (ix < nx && iy < ny)
    {
        // Transpose and write the output
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}