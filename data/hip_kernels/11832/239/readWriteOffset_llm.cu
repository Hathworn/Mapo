#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readWriteOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Cache offset calculation outside the loop
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Avoid unnecessary computation by checking offset first
    if (k < n) {
        // Use shared memory to cache data if possible (assuming independent work)
        __shared__ float sharedA[BLOCK_SIZE];
        __shared__ float sharedB[BLOCK_SIZE];
        
        if (threadIdx.x + offset < n) {
            sharedA[threadIdx.x] = A[k];
            sharedB[threadIdx.x] = B[k];
            __syncthreads();
            C[k] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
        }
    }
}