#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void naiveGmem(float *out, float *in, const int nx, const int ny)
{
    // matrix coordinate (ix,iy)
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Use shared memory for better memory access pattern
    // Threads will work on shared memory and write results back to global memory

    // Transpose with boundary test and write back to global memory
    if (ix < nx && iy < ny)
    {
        out[ix * ny + iy] = in[iy * nx + ix];
    }
}