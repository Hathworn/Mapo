#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling8(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8 with register caching for reduced memory latency
    int sum = 0;
    if (idx + 7 * blockDim.x < n)
    {
        // accumulate sum using shared memory
        sum += g_idata[idx];
        sum += g_idata[idx + blockDim.x];
        sum += g_idata[idx + 2 * blockDim.x];
        sum += g_idata[idx + 3 * blockDim.x];
        sum += g_idata[idx + 4 * blockDim.x];
        sum += g_idata[idx + 5 * blockDim.x];
        sum += g_idata[idx + 6 * blockDim.x];
        sum += g_idata[idx + 7 * blockDim.x];

        g_idata[idx] = sum;
    }

    __syncthreads();

    // in-place reduction in global memory using shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0)
    {
        g_odata[blockIdx.x] = idata[0];
    }
}