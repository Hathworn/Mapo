#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffsetUnroll4(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    unsigned int k = i + offset;

    // Use loop unrolling only when within bounds
    if (k < n)
    {
        #pragma unroll 4
        for (int j = 0; j < 4; j++)
        {
            if (k + j * blockDim.x < n)
            {
                C[i + j * blockDim.x] = A[k + j * blockDim.x] + B[k + j * blockDim.x];
            }
        }
    }
}