#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighboredSmem(int *g_idata, int *g_odata, unsigned int  n)
{
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n) return;

    // load data to shared memory
    smem[tid] = g_idata[idx];
    __syncthreads();

    // optimized in-place reduction with shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            smem[tid] += smem[tid + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}