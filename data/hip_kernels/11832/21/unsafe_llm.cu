#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unsafe(int *shared_var, int *values_read, int N, int iters)
{
    int i;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N) return;

    // Use atomicAdd to safely update shared_var in a concurrent environment
    int old = atomicAdd(shared_var, 1);
    values_read[tid] = old;

    // Loop with atomicAdd to ensure thread-safe updates to shared_var
    for (i = 0; i < iters; i++)
    {
        atomicAdd(shared_var, 1);
    }
}