#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceCompleteUnrollWarps8 (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    
    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;
    
    // Unrolling 8 and boundary check
    if (idx + 7 * blockDim.x < n)
    {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        int b1 = g_idata[idx + 4 * blockDim.x];
        int b2 = g_idata[idx + 5 * blockDim.x];
        int b3 = g_idata[idx + 6 * blockDim.x];
        int b4 = g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = a1 + a2 + a3 + a4 + b1 + b2 + b3 + b4;
    }
    
    __syncthreads();

    // In-place reduction with manual unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();
    
    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();
    
    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();
    
    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();
    
    // Unrolling warp using volatile memory to prevent optimization issues
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }
    
    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}