#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global index with offset applied
    unsigned int k = blockIdx.x * blockDim.x + threadIdx.x + offset;
    
    // Ensure the computed index is within bounds
    if (k < n) {
        C[k - offset] = A[k] + B[k]; // Use C[k - offset] to align with input arrays
    }
}