#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceInterleaved(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if (idx >= n) return;

    // In-place reduction in global memory
    // Optimized: Load data into shared memory (coalesced access)
    extern __shared__ int smem[];
    smem[tid] = idata[tid];
    __syncthreads();

    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            smem[tid] += smem[tid + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}