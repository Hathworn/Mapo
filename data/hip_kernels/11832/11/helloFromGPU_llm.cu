#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU()
{
    // Load thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure only one thread executes the print statement
    if (tid == 0) {
        printf("Hello World from GPU!\n");
    }
}