#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n) {
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int* idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check; return early if out of range
    if (idx >= n) return;

    // In-place reduction using shared memory
    __shared__ int shared_idata[blockDim.x];
    shared_idata[tid] = g_idata[idx];
    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        int index = 2 * stride * tid;
        if (index < blockDim.x) {
            shared_idata[index] += shared_idata[index + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = shared_idata[0];
}