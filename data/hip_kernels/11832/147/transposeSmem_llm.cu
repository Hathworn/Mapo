#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_DIM 16 // Assuming square block size for simplicity

__global__ void transposeSmem(float *out, float *in, int nx, int ny)
{
    // Static shared memory with adjusted dimensions for memory coalescing
    __shared__ float tile[BLOCK_DIM][BLOCK_DIM + 1];

    // Original matrix coordinates
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny) {
        // Linear global memory index for original matrix
        unsigned int ti = iy * nx + ix;

        // Load data from global memory to shared memory
        tile[threadIdx.y][threadIdx.x] = in[ti];
    }

    // Thread synchronization
    __syncthreads();

    // Coordinates for the transposed matrix
    ix = blockDim.y * blockIdx.y + threadIdx.x;
    iy = blockDim.x * blockIdx.x + threadIdx.y;

    if (ix < ny && iy < nx) {
        // Linear global memory index for transposed matrix
        unsigned int to = iy * ny + ix;

        // Store data to global memory from shared memory with transposed index
        out[to] = tile[threadIdx.x][threadIdx.y];
    }
}