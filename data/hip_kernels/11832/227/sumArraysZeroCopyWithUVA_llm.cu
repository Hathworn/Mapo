#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysZeroCopyWithUVA(float *A, float *B, float *C, const int N)
{
    // Calculate unique thread index efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread id is within bounds before accessing arrays
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}