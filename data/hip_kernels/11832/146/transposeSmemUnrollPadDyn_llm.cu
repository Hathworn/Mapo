#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeSmemUnrollPadDyn(float *out, float *in, const int nx, const int ny) {
    // Use dynamic shared memory
    extern __shared__ float tile[];

    unsigned int ix = blockDim.x * blockIdx.x * 2 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = bidx / blockDim.y;
    unsigned int icol = bidx % blockDim.y;

    // Calculate transposed matrix coordinates
    unsigned int ix2 = blockDim.y * blockIdx.y + icol;
    unsigned int iy2 = blockDim.x * 2 * blockIdx.x + irow;
    unsigned int to = iy2 * ny + ix2;

    // Transpose with boundary test
    if (ix + blockDim.x < nx && iy < ny) {
        // Load data from global memory to shared memory
        unsigned int row_idx = threadIdx.y * (blockDim.x * 2 + IPAD) + threadIdx.x;
        tile[row_idx] = in[iy * nx + ix];
        tile[row_idx + BDIMX] = in[iy * nx + ix + BDIMX];

        // Synchronize threads
        __syncthreads();

        unsigned int col_idx = icol * (blockDim.x * 2 + IPAD) + irow;
        out[to] = tile[col_idx];
        out[to + ny * BDIMX] = tile[col_idx + BDIMX];
    }
}