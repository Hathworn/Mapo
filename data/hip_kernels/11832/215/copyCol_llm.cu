#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate global linear index
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int index = ix * ny + iy;

    // Use a single if-statement for combined bounds checking
    if (ix < nx && iy < ny)
    {
        out[index] = in[index];
    }
}