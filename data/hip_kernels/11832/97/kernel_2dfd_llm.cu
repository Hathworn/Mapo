#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_2dfd(float *g_u1, float *g_u2, const int nx, const int iStart, const int iEnd)
{
    // global to line index
    unsigned int ix  = blockIdx.x * blockDim.x + threadIdx.x;

    // shared memory for x dimension
    __shared__ float line[BDIMX + NPAD2];

    // a coefficient related to physical properties
    const float alpha = 0.12f;

    // register for y value
    float yval[9];

    // Initialize y values
    for (int i = 0; i < 8; i++) {
        unsigned int idx  = ix + (iStart + i - 4) * nx;
        yval[i] = g_u2[idx];
    }

    // loop over the y dimension
    for (int iy = iStart; iy < iEnd; iy++)
    {
        // Calculate global idx for current y-level
        unsigned int idx = ix + iy * nx;

        // Load yval[8] asynchronously
        yval[8] = g_u2[idx + (4 * nx)];

        // read halo part
        if (threadIdx.x < NPAD) {
            line[threadIdx.x]  = g_u2[idx - NPAD];
            line[stx + BDIMX]  = g_u2[idx + BDIMX];
        }

        // load the primary argument 
        line[stx] = yval[4];
        __syncthreads();

        // 8th order finite difference operator
        if (ix >= NPAD && ix < nx - NPAD) {
            float tmp = coef[0] * line[stx] * 2.0f;

            #pragma unroll
            for (int d = 1; d <= 4; d++) {
                tmp += coef[d] * (line[stx - d] + line[stx + d]);
            }

            #pragma unroll
            for (int d = 1; d <= 4; d++) {
                tmp += coef[d] * (yval[4 - d] + yval[4 + d]);
            }
            
            // Update output with new values
            g_u1[idx] = yval[4] + yval[4] - g_u1[idx] + alpha * tmp;
        }

        // shift the array yval for next computation
        #pragma unroll
        for (int i = 0; i < 8; i++) {
            yval[i] = yval[i + 1];
        }

        __syncthreads(); // Ensure all threads have completed before proceeding
    }
}