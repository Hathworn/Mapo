#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_add_wavelet(float *g_u2, float wavelets, const int nx, const int ny, const int ngpus)
{
    // Compute global thread index
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit if the index is out of bounds
    if (ix >= nx) return;

    // Calculate index specific to 2 GPUs setup or single GPU setup
    int ipos = (ngpus == 2 ? ny - 10 : ny / 2 - 10);
    unsigned int idx = ipos * nx + ix;

    // Perform the wavelet addition only for the specified index
    if (ix == nx / 2) {
        g_u2[idx] += wavelets;
    }
}