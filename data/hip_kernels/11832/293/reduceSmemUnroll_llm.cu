#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // Static shared memory
    __shared__ int smem[DIM];

    // Set thread ID and global index, 4 blocks of input data processed at a time
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // Unrolling 4 blocks with boundary check
    int tmpSum = (idx + 3 * blockDim.x < n) ? g_idata[idx] + g_idata[idx + blockDim.x] +
                g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x] : 0;

    smem[tid] = tmpSum;
    __syncthreads();

    // In-place reduction in shared memory
    if (blockDim.x >= 1024) { if (tid < 512) smem[tid] += smem[tid + 512]; __syncthreads(); }
    if (blockDim.x >= 512)  { if (tid < 256) smem[tid] += smem[tid + 256]; __syncthreads(); }
    if (blockDim.x >= 256)  { if (tid < 128) smem[tid] += smem[tid + 128]; __syncthreads(); }
    if (blockDim.x >= 128)  { if (tid <  64) smem[tid] += smem[tid +  64]; __syncthreads(); }

    // Unrolling warp with volatile memory
    if (tid < 32)
    {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}