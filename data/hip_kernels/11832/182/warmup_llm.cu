#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(int *out, int N) {
    // Utilize shared memory for the computation
    extern __shared__ int sharedOut[];

    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
        sharedOut[threadIdx.x] = 0;
        __syncthreads();
        out[tid] = sharedOut[threadIdx.x];
    }
}