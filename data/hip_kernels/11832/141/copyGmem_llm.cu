#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyGmem(float *out, float *in, const int nx, const int ny)
{
    // matrix coordinate (ix,iy)
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // transpose with boundary check
    if (ix < nx && iy < ny)
    {
        // Optimized memory access pattern
        out[iy * nx + ix] = in[iy * nx + ix];
    }
}