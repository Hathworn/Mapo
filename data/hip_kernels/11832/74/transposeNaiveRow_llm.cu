#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNaiveRow(float *out, float *in, const int nx, const int ny)
{
    // Calculate global thread position
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure indices are within bounds using shared memory (coalescing memory access)
    __shared__ float tile[32][33]; // Adjust tile size to handle bank conflicts

    if (ix < nx && iy < ny)
    {
        // Load data to shared memory
        tile[threadIdx.y][threadIdx.x] = in[iy * nx + ix];
        __syncthreads();

        // Write data to output in transposed order
        out[ix * ny + iy] = tile[threadIdx.y][threadIdx.x];
    }
}