#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readOffsetUnroll2(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;
    unsigned int k = i + offset;

    // Optimize by reducing conditions and simplifying index calculations
    if (k < n)
    {
        C[i] = A[k] + B[k];
        if (k + blockDim.x < n) // Check boundary before the second assignment
        {
            C[i + blockDim.x] = A[k + blockDim.x] + B[k + blockDim.x];
        }
    }
}