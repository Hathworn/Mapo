#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void device_api_kernel(hiprandState *states, float *out, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;

    if (tid < N)
    {
        // Initialize the state only if the thread processes some elements
        hiprandState state;
        hiprand_init(9384, tid, 0, &state);

        // Use a temporary state variable to improve memory access
        for (int i = tid; i < N; i += nthreads)
        {
            float rand = hiprand_uniform(&state) * 2;
            out[i] = rand;
        }
    }
}