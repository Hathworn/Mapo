#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iKernel(float *src, float *dst)
{
    // Utilize shared memory for improved memory access efficiency
    extern __shared__ float shared_src[];
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data into shared memory
    shared_src[threadIdx.x] = src[idx];
    __syncthreads();
    
    // Perform computation using shared memory
    dst[idx] = shared_src[threadIdx.x] * 2.0f;
}