#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeSmemPadDyn(float *out, float *in, int nx, int ny)
{
    // Dynamically allocated shared memory with padding
    extern __shared__ float tile[];

    // Coordinate in the original matrix calculated by each thread
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    unsigned int ti = iy * nx + ix; // Linear global memory index

    // Thread index calculations for the transposed block
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int row_idx = threadIdx.y * (blockDim.x + IPAD) + threadIdx.x;
    unsigned int irow = idx / blockDim.y;
    unsigned int icol = idx % blockDim.y;
    unsigned int col_idx = icol * (blockDim.x + IPAD) + irow;

    // Coordinate in the transposed matrix
    unsigned int t_ix = blockDim.y * blockIdx.y + icol;
    unsigned int t_iy = blockDim.x * blockIdx.x + irow;

    // Linear global memory index for transposed matrix
    unsigned int to = t_iy * ny + t_ix;

    // Transpose with boundary test
    if (ix < nx && iy < ny) 
    {
        // Load data from global memory to shared memory
        tile[row_idx] = in[ti];
    }

    // Thread synchronization
    __syncthreads();

    if (t_ix < ny && t_iy < nx) 
    {
        // Store data to global memory from shared memory
        out[to] = tile[col_idx];
    }
}