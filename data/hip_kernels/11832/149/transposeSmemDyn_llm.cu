#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeSmemDyn(float *out, float *in, int nx, int ny)
{
    // Dynamic shared memory allocation
    extern __shared__ float tile[];

    // Calculate index for input matrix
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Linear index for global memory in input matrix
    if (ix < nx && iy < ny)
    {
        unsigned int ti = iy * nx + ix;

        // Reorder thread index for transposed access
        unsigned int row_idx = threadIdx.y * blockDim.x + threadIdx.x;
        unsigned int irow    = row_idx / blockDim.y;
        unsigned int icol    = row_idx % blockDim.y;
        unsigned int col_idx = icol * blockDim.x + irow;

        // Calculate index for output(transposed) matrix
        ix = blockDim.y * blockIdx.y + icol;
        iy = blockDim.x * blockIdx.x + irow;

        // Load data from global memory to shared memory
        tile[row_idx] = in[ti];

        // Synchronize to ensure all data is loaded
        __syncthreads();

        // Linear index for global memory in output matrix
        if (ix < nx && iy < ny)
        {
            unsigned int to = iy * ny + ix;
            // Store data from shared memory to global memory
            out[to] = tile[col_idx];
        }
    }
}