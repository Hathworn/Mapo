#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iKernel(float *src, float *dst, int dataSize)
{
    // Compute global index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we're within the bounds of the input data
    if (idx < dataSize) {
        dst[idx] = src[idx] * 2.0f;
    }
}