#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyRow(float *out, float *in, const int nx, const int ny)
{
    // Efficiently calculate row-major index in 2D grid
    unsigned int rowIdx = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int colIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (rowIdx < ny && colIdx < nx)
    {
        unsigned int index = rowIdx * nx + colIdx; // Compute 1D index for 2D data
        out[index] = in[index];
    }
}