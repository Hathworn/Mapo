#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    // Calculate thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check index bounds
    if (idx < N)
    {
        // Sum arrays element-wise
        C[idx] = A[idx] + B[idx];
    }
}