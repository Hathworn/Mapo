#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(float *A, float *B, float *C, const int N)
{
    // Use shared memory to optimize data access
    __shared__ float s_A[1024];
    __shared__ float s_B[1024];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < N) {
        // Load data into shared memory
        s_A[threadIdx.x] = A[i];
        s_B[threadIdx.x] = B[i];
        __syncthreads();

        // Perform addition using shared memory
        C[i] = s_A[threadIdx.x] + s_B[threadIdx.x];
    }
}