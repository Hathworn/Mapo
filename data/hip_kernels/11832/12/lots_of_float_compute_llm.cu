#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void lots_of_float_compute(float *inputs, int N, size_t niters, float *outputs)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t nthreads = gridDim.x * blockDim.x;

    for ( ; tid < N; tid += nthreads)
    {
        float val = inputs[tid];

        // Combine constant operations outside the loop to reduce computation
        float const1 = (5.0f - 101.0f) / 3.0f + 102.0f;
        float const2 = (1.07f - 103.0f) / 1.037f + 104.0f;
        float const3 = (3.00f - 105.0f) / 0.22f + 106.0f;

        for (size_t iter = 0; iter < niters; iter++)
        {
            val = const1;
            val = const2;
            val = const3;
        }

        outputs[tid] = val;
    }
}