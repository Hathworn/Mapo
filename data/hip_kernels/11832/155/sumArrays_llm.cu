#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Execute loop conditionally to eliminate unnecessary iterations
    if (idx < N)
    {
        // Perform the addition operation directly for this index
        C[idx] = A[idx] + B[idx];
    }
}