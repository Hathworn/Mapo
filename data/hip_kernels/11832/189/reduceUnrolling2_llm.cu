#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling2 (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // unrolling 2
    int sum = 0;
    if (idx < n) sum = g_idata[idx];
    if (idx + blockDim.x < n) sum += g_idata[idx + blockDim.x];

    // in-place reduction in shared memory
    __shared__ int smem[512];  // Assuming blockDim.x <= 512
    smem[tid] = sum;
    __syncthreads();

    // unrolled loop for reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            smem[tid] += smem[tid + stride];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}