#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeDiagonalCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate block and thread indices
    unsigned int blk_y = blockIdx.x;
    unsigned int blk_x = (blockIdx.x + blockIdx.y) % gridDim.x;

    // Calculate global coordinates and check bounds
    unsigned int ix = blockDim.x * blk_x + threadIdx.x;
    unsigned int iy = blockDim.y * blk_y + threadIdx.y;

    // Use shared memory to optimize memory access patterns
    __shared__ float tile[BLOCK_SIZE][BLOCK_SIZE + 1]; // +1 to avoid bank conflict

    if (ix < nx && iy < ny)
    {
        tile[threadIdx.y][threadIdx.x] = in[ix * ny + iy]; // Load data into shared memory
    }
    __syncthreads();

    ix = blockDim.y * blk_y + threadIdx.x; // Transpose coordinates
    iy = blockDim.x * blk_x + threadIdx.y;

    if (ix < ny && iy < nx)
    {
        out[iy * nx + ix] = tile[threadIdx.x][threadIdx.y]; // Write transposed data from shared memory
    }
}