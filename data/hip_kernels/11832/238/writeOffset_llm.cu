#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Ensure index within bounds and perform computation
    if (k < n)
    {
        C[k] = A[i] + B[i];
    }
}