#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceGmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    
    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 4;
    
    // unrolling 4
    if (idx + 3 * blockDim.x < n)
    {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        idata[tid] = a1 + a2 + a3 + a4;  // Combine results into idata
    }
    
    __syncthreads();
    
    // Optimize reduction using shared memory
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();
    
    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();
    
    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();
    
    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();
    
    // unrolling warp using shared memory
    if (tid < 32)
    {
        volatile int *smem = idata;  // Use shared memory for warp-level reductions
        smem[tid] += smem[tid + 32];
        smem[tid] += smem[tid + 16];
        smem[tid] += smem[tid +  8];
        smem[tid] += smem[tid +  4];
        smem[tid] += smem[tid +  2];
        smem[tid] += smem[tid +  1];
    }
    
    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}