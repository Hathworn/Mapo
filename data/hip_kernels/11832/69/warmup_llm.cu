#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate the global index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Ensure we do not access out-of-bounds memory
    if (k < n) {
        C[i] = A[k] + B[k];
    }
}