#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    // Obtain global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop to process multiple elements per thread for larger workloads
    for (int idx = i; idx < N; idx += gridDim.x * blockDim.x)
    {
        C[idx] = A[idx] + B[idx];
    }
}