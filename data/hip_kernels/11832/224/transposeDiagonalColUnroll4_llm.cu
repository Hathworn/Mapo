#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalColUnroll4(float *out, float *in, const int nx, const int ny)
{
    unsigned int blk_idx = blockIdx.x + blockIdx.y * gridDim.x;
    unsigned int ix_stride = blockDim.x * gridDim.x;
    unsigned int ix = blk_idx * blockDim.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Ensure thread index calculations do not exceed boundaries
    if (iy < ny)
    {
        if (ix < nx) out[iy * nx + ix] = in[ix * ny + iy];
        if (ix + blockDim.x < nx) out[iy * nx + ix + blockDim.x] = in[(ix + blockDim.x) * ny + iy];
        if (ix + 2 * blockDim.x < nx) out[iy * nx + ix + 2 * blockDim.x] = in[(ix + 2 * blockDim.x) * ny + iy];
        if (ix + 3 * blockDim.x < nx) out[iy * nx + ix + 3 * blockDim.x] = in[(ix + 3 * blockDim.x) * ny + iy];
    }
}