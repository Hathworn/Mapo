#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y; // Use 2D grid and block
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny)
        MatC[idx] = MatA[idx] + MatB[idx];
}