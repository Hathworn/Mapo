#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void iKernel(float *A, float *B, float *C, const int N)
{
    // Use shared memory for better performance
    extern __shared__ float shared_mem[];
    float *shared_A = shared_mem;
    float *shared_B = shared_mem + blockDim.x;

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        // Load data into shared memory
        shared_A[threadIdx.x] = A[i];
        shared_B[threadIdx.x] = B[i];
        __syncthreads();

        // Perform computation using shared memory
        C[i] = shared_A[threadIdx.x] + shared_B[threadIdx.x];
    }
}