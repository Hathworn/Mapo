#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iKernel(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use collective memory access for better performance
    if (i < N)
    {
        const float a = __ldg(&A[i]);
        const float b = __ldg(&B[i]);
        C[i] = a + b;
    }
}