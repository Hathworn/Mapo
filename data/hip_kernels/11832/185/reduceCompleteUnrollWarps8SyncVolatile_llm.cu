#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceCompleteUnrollWarps8SyncVolatile(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8
    if (idx + 7 * blockDim.x < n)
    {
        int sum = 0;
        sum += g_idata[idx];
        sum += g_idata[idx + blockDim.x];
        sum += g_idata[idx + 2 * blockDim.x];
        sum += g_idata[idx + 3 * blockDim.x];
        sum += g_idata[idx + 4 * blockDim.x];
        sum += g_idata[idx + 5 * blockDim.x];
        sum += g_idata[idx + 6 * blockDim.x];
        sum += g_idata[idx + 7 * blockDim.x];
        g_idata[idx] = sum;
    }
    __syncthreads();

    // in-place reduction and complete unroll
    if (blockDim.x >= 1024 && tid < 512) idata[tid] += idata[tid + 512];
    __syncthreads();
    if (blockDim.x >= 512 && tid < 256) idata[tid] += idata[tid + 256];
    __syncthreads();
    if (blockDim.x >= 256 && tid < 128) idata[tid] += idata[tid + 128];
    __syncthreads();
    if (blockDim.x >= 128 && tid < 64) idata[tid] += idata[tid + 64];
    __syncthreads();

    volatile int *vmem = idata;
    // unrolling warp reduction, remove unnecessary barriers within warp
    if (tid < 32)
    {
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }
    
    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}