#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaiveCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate transposed global indices
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Use shared memory for more efficient memory access
    __shared__ float tile[32][32 + 1]; // Adding padding to avoid bank conflicts

    if (ix < nx && iy < ny)
    {
        unsigned int index_in = iy * nx + ix;
        tile[threadIdx.y][threadIdx.x] = in[index_in];
    }
    
    __syncthreads();

    ix = blockDim.y * blockIdx.y + threadIdx.x; // Transpose - swap indices
    iy = blockDim.x * blockIdx.x + threadIdx.y;
    
    if (iy < nx && ix < ny)
    {
        unsigned int index_out = ix * nx + iy;
        out[index_out] = tile[threadIdx.x][threadIdx.y];
    }
}