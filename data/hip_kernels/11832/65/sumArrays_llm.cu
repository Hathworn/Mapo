#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use if statement to check boundary, unnecessary calculations avoided
    if (i < N) 
    {
        C[i] = A[i] + B[i];
    }
}