#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void intrinsic_kernel(float a, float *out, int iters)
{
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;
    // Reduce thread divergence, use shared memory for optimization
    __shared__ float tmp;
    
    if(tid == 0)
    {
        tmp = __powf(a, 2.0f);
    }
    
    __syncthreads();
    
    if(tid == 0)
    {
        *out = tmp; // Single assignment after synchronization
    }
}