#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refill_randoms(float *dRand, int N, hiprandState *states)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x; // Simplified total threads calculation
    hiprandState localState = states[tid]; // Load the state into a local variable

    for (int i = tid; i < N; i += stride)
    {
        dRand[i] = hiprand_uniform(&localState); // Use the local state variable
    }

    states[tid] = localState; // Store the updated state back to global memory
}