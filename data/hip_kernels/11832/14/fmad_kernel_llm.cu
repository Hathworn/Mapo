#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fmad_kernel(double x, double y, double *out)
{
    // Use shared memory to store results, reducing global memory access
    __shared__ double result;

    int tid = threadIdx.x;
    if (tid == 0)
    {
        // Perform operation and store in shared memory
        result = x * x + y;
    }

    // Ensure all threads have computed and written to shared memory
    __syncthreads();

    if (tid == 0)
    {
        // Write the result from shared memory to the global memory
        *out = result;
    }
}