#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, int NX, int NY)
{
    // Use shared memory for faster access
    extern __shared__ float sData[];

    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * NX + ix;

    if (ix < NX && iy < NY)
    {
        // Load data into shared memory
        sData[threadIdx.y * blockDim.x + threadIdx.x] = A[idx] + B[idx];

        // Ensure all threads have written their data
        __syncthreads();

        // Write the result back to global memory
        C[idx] = sData[threadIdx.y * blockDim.x + threadIdx.x];
    }
}