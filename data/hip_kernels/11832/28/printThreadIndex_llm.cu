#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void printThreadIndex(int *A, const int nx, const int ny)
{
    // Precompute the index for improved readability and maintenance.
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Calculate the linear index in the array.
    unsigned int idx = iy * nx + ix;

    // Use a single printf for better performance.
    printf("thread_id (%d,%d) block_id (%d,%d) coordinate (%d,%d) global index %2d ival %2d\n", 
           threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx, A[idx]);
}