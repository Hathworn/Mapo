#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *out, float *in, const int nx, const int ny)
{
    // Calculate global thread index
    unsigned int idx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int idy = blockDim.y * blockIdx.y + threadIdx.y;

    // Use a direct index calculation
    unsigned int index = idy * nx + idx;

    // Prevent out-of-bounds access
    if (idx < nx && idy < ny)
    {
        out[index] = in[index];
    }
}