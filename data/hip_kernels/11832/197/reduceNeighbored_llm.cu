#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // exit if thread index is beyond the range
    if (idx >= n) return;

    // optimize: load data into shared memory to minimize global memory access
    extern __shared__ int sdata[];
    sdata[tid] = g_idata[idx];
    __syncthreads();

    // in-place reduction in shared memory
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        if (tid % (2 * stride) == 0)
        {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads(); // synchronize within threadblock
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}