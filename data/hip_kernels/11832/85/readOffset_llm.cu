#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate the global thread index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Use a direct array index check to improve efficiency
    if (i < n - offset) 
    {
        C[i] = A[k] + B[k];
    }
}