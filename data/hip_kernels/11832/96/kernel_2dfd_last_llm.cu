#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_2dfd_last(float *g_u1, float *g_u2, const int nx, const int iStart, const int iEnd)
{
    unsigned int ix  = blockIdx.x * blockDim.x + threadIdx.x;

    // Shared memory optimization
    __shared__ float tile[BDIMX + NPAD2];

    // Optimized register usage for yval
    float yval[9];

    const float alpha = 0.12f;
    unsigned int idx  = ix + iStart * nx;
    unsigned int stx = threadIdx.x + NPAD;
    int iskip = NPAD * nx;

    // Load initial values into registers
    for (int i = 0; i < 8; i++) {
        yval[i] = g_u2[idx + (i - 4) * nx];
    }

    // Loop over y dimension
    #pragma unroll 9
    for (int iy = iStart; iy < iEnd; iy++)
    {
        // Prefetch next yval[8] into register
        yval[8] = g_u2[idx + iskip];

        // Load left and right halo into shared memory
        if(threadIdx.x < NPAD)
        {
            tile[threadIdx.x] = g_u2[idx - NPAD];
            tile[stx + BDIMX] = g_u2[idx + BDIMX];
        }

        tile[stx] = yval[4];
        __syncthreads();

        if ((ix >= NPAD) && (ix < nx - NPAD))
        {
            float tmp = coef[0] * tile[stx] * 2.0f;

            // Unroll loop for x-dimension finite difference computation
            #pragma unroll
            for(int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (tile[stx - d] + tile[stx + d]);
            }

            // Unroll loop for y-dimension finite difference computation
            #pragma unroll
            for(int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (yval[4 - d] + yval[4 + d]);
            }

            // Update global memory
            g_u1[idx] = yval[4] + yval[4] - g_u1[idx] + alpha * tmp;
        }

        // Shift y values for next iteration
        #pragma unroll 8
        for (int i = 0; i < 8; i++)
        {
            yval[i] = yval[i + 1];
        }

        // Advance to next index in global memory
        idx += nx;
        __syncthreads();
    }
}