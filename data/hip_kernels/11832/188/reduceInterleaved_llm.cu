#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceInterleaved (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if (idx >= n) return;

    // In-place reduction in shared memory for efficient collaboration between threads
    extern __shared__ int sdata[];
    sdata[tid] = idata[tid];
    __syncthreads();

    // Perform reduction using shared memory for coalesced memory access
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}