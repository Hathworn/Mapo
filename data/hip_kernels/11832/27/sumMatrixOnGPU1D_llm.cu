#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Use 2D grid and block to handle 2D matrix operation more efficiently
    if (ix < nx && iy < ny)
    {
        int idx = iy * nx + ix;
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}