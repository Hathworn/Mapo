#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x; // Compute global x index
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y; // Compute global y index

    if (ix < nx && iy < ny) // Ensure indices are within bounds
    {
        unsigned int idx = iy * nx + ix; // Linearized index calculation
        MatC[idx] = MatA[idx] + MatB[idx]; // Perform matrix addition
    }
}