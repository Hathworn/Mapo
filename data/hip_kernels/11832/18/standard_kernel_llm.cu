#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void standard_kernel(float a, float *out, int iters)
{
    int tid = (blockDim.x * blockIdx.x) + threadIdx.x;

    // Utilize shared memory to potentially speed up memory access
    __shared__ float shared_tmp;

    if(tid == 0)
    {
        // Perform calculation only once, since result doesn't change across iterations
        shared_tmp = powf(a, 2.0f);

        // Ensure all threads complete the calculation
        __syncthreads();

        if (iters > 0)
        {
            *out = shared_tmp;
        }
    }
}