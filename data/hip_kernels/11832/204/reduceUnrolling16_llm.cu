#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling16 (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 16 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 16;

    // unrolling 16
    if (idx + 15 * blockDim.x < n)
    {
        int sum = 0;
        #pragma unroll
        for (int i = 0; i < 16; i++) {
            sum += g_idata[idx + i * blockDim.x];
        }
        g_idata[idx] = sum;
    }
    __syncthreads();

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}