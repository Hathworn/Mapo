#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysZeroCopy(float *A, float *B, float *C, const int N)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use if statement outside the loop to avoid branching inside the loop
    if (i < N) {
        // Perform computation
        C[i] = A[i] + B[i];
    }
}