#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int k = blockIdx.x * blockDim.x + threadIdx.x + offset; // Combine offset addition

    if (k < n) {
        C[k] = A[k - offset] + B[k - offset]; // Use k for indexing with adjusted offset
    }
}