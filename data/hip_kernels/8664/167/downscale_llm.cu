#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    // Calculate index inverse with scaling and offset
    int x = ii / (d1 * d2 * d3);
    int y = (ii / (d2 * d3)) % d1;
    int z = (ii / d3) % d2;
    int w = ii % d3;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    // Calculate index forward with scaling
    int x = ii / (d1 * d2 * d3);
    int y = (ii / (d2 * d3)) % d1;
    int z = (ii / d3) % d2;
    int w = ii % d3;
    w = w / scale_factor;
    z = z / scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    // Calculate unique thread index for 2D thread blocks
    long ii = threadIdx.x + blockDim.x * blockIdx.x +
              (threadIdx.y + blockDim.y * blockIdx.y) * blockDim.x * gridDim.x;
              
    // Return if index is out of bounds
    if (ii >= no_elements) return;

    // Use loop unrolling to reduce loop overhead
    for (int i = 0; i < scale_factor; ++i) {
        for (int j = 0; j < scale_factor; ++j) {
            int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
            gradInput_data[ii] += gradOutput_data[ipidx];
        }
    }
}