#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate global index
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    int i = blockDim.y * blockIdx.y + threadIdx.y;

    // Use early exit for threads outside of bounds
    if (i < h && j < w) {
        int pos = i * w + j; // Calculate linear index
        image[pos] = value; // Set value
    }
}