#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Use shared memory for coalesced access if beneficial
    __shared__ short2 shared_loc[256]; // Adjust size based on blockDim.x and occupancy 

    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        shared_loc[threadIdx.x] = loc_[ptidx]; // Load from global to shared memory
        __syncthreads(); // Ensure all threads have loaded the data

        const short2 loc = shared_loc[threadIdx.x]; // Access shared memory
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}