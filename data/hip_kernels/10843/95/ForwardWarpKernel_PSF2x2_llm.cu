#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float* u, const float* v, const float* src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float* normalization_factor, float* dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Use shared memory for improved access speed
    extern __shared__ float shared_src[];
    shared_src[threadIdx.y * blockDim.x + threadIdx.x] = src[image_row_offset + j];
    __syncthreads();

    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    int tx = (int)px;
    int ty = (int)py;

    float value = shared_src[threadIdx.y * blockDim.x + threadIdx.x];
    float weight;

    // Macro to avoid repetitive checks
    #define IS_VALID_PIXEL(tx, ty) (!(tx >= w || tx < 0 || ty >= h || ty < 0))

    if (IS_VALID_PIXEL(tx, ty))
    {
        weight = dx * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    tx -= 1;
    if (IS_VALID_PIXEL(tx, ty))
    {
        weight = (1.0f - dx) * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    ty -= 1;
    if (IS_VALID_PIXEL(tx, ty))
    {
        weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    tx += 1;
    if (IS_VALID_PIXEL(tx, ty))
    {
        weight = dx * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    #undef IS_VALID_PIXEL
}