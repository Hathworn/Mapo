#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(double *a, double *b, double *c, int n) 
{
    // Calculate the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a stride loop for grid-stride access pattern
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < n; i += stride) {
        c[i] = a[i] + b[i];
    }
}