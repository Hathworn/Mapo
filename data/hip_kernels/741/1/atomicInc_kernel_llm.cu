#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define VERBOSE 0
#define INTEGER_SCALE_FACTOR 100

// Command line argument definitions
#define DEFAULT_NUM_REPEATS 1
#define DEFAULT_NUM_ITERATIONS 1
#define DEFAULT_NUM_ELEMENTS 128
#define DEFAULT_SEED 0
#define DEFAULT_DEVICE 0

#define MIN_ARGS 1
#define MAX_ARGS 6

#define ARG_EXECUTABLE 0
#define ARG_REPEATS 1
#define ARG_ITERATIONS 2
#define ARG_ELEMENTS 3
#define ARG_SEED 4
#define ARG_DEVICE 5

#define MAX 10

// Optimized CUDA Kernel
__global__ void atomicInc_kernel( unsigned int numIterations, unsigned int numInputs, float * d_probabilities, unsigned int * d_quantity, unsigned int * d_count ){
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread id is within the bounds
    if (tid >= numInputs) return;

    // Remove if checks, consolidate printf 
    printf("tid %u: d_quantity[%u] = %u\n", tid, tid, d_quantity[tid]);

    for (int iteration = 0; iteration < numIterations; iteration++){
        // Atomic increment with clamp
        unsigned int old = atomicInc(d_quantity + tid, MAX);

        // Increment count if old < MAX
        if (old < MAX) {
            d_count[tid]++;
        }
    }
}