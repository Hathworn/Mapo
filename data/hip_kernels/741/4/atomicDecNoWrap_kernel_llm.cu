#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define VERBOSE 0
#define INTEGER_SCALE_FACTOR 100

// Command line argument definitions
#define DEFAULT_NUM_REPEATS 1
#define DEFAULT_NUM_ITERATIONS 1
#define DEFAULT_NUM_ELEMENTS 128
#define DEFAULT_SEED 0
#define DEFAULT_DEVICE 0

#define MIN_ARGS 1
#define MAX_ARGS 6

#define ARG_EXECUTABLE 0
#define ARG_REPEATS 1
#define ARG_ITERATIONS 2
#define ARG_ELEMENTS 3
#define ARG_SEED 4
#define ARG_DEVICE 5

#define MAX 10

// Lazy CUDA Error handling
__device__ unsigned int atomicDecNoWrap(unsigned int * address, unsigned int val) {
    unsigned int old = *address;
    unsigned int assumed;
    do {
        assumed = old;
        old = atomicCAS(address, assumed, (((assumed == 0) || (assumed > val)) ? assumed : (assumed - 1)));
    } while (assumed != old);
    return old;
}

__global__ void atomicDecNoWrap_kernel(unsigned int numIterations, unsigned int numInputs, float * d_probabilities, 
                                       unsigned int * d_quantity, unsigned int * d_count) {
    unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);

    // Ensure thread is within range
    if (tid < numInputs) {
        // Print the initial quantity for first thread
        if (tid == 0) {
            printf("d_quantity[%u] = %u\n", tid, d_quantity[tid]);
        }

        // Loop through iterations
        for (int iteration = 0; iteration < numIterations; iteration++) {
            // Atomic decrement
            unsigned int old = atomicDecNoWrap(d_quantity + tid, MAX);

            // Print the old value for first thread on each iteration
            if (tid == 0) {
                printf("tid %u: iter %d, old %u\n", tid, iteration, old);
            }

            // If old is greater than 0, increment count
            if (old > 0) {
                atomicAdd(d_count + tid, 1); // Use atomicAdd for safe increments
            }
        }
    }
}