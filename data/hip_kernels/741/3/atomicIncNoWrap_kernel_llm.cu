#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define VERBOSE 0
#define INTEGER_SCALE_FACTOR 100

#define DEFAULT_NUM_REPEATS 1
#define DEFAULT_NUM_ITERATIONS 1
#define DEFAULT_NUM_ELEMENTS 128
#define DEFAULT_SEED 0
#define DEFAULT_DEVICE 0

#define MIN_ARGS 1
#define MAX_ARGS 6

#define ARG_EXECUTABLE 0
#define ARG_REPEATS 1
#define ARG_ITERATIONS 2
#define ARG_ELEMENTS 3
#define ARG_SEED 4
#define ARG_DEVICE 5

#define MAX 10

__device__ unsigned int atomicIncNoWrap(unsigned int *address, unsigned int val) {
    unsigned int old = *address;
    unsigned int assumed;
    do {
        assumed = old;
        old = atomicCAS(address, assumed, ((assumed >= val) ? assumed : (assumed + 1)));
    } while (assumed != old);
    return old;
}

__global__ void atomicIncNoWrap_kernel(unsigned int numIterations, unsigned int numInputs, float *d_probabilities, unsigned int *d_quantity, unsigned int *d_count) {
    unsigned int tid = threadIdx.x + (blockDim.x * blockIdx.x);

    // Use shared memory for d_count to reduce global memory accesses
    extern __shared__ unsigned int shared_count[];

    if (tid < numInputs) {
        if (tid == 0) {
            printf("d_quantity[%u] = %u\n", tid, d_quantity[tid]);
        }
        // Initialize shared memory
        shared_count[threadIdx.x] = 0;

        for (int iteration = 0; iteration < numIterations; iteration++) {
            unsigned int old = atomicIncNoWrap(d_quantity + tid, MAX);
            if (tid == 0) {
                printf("tid %u: iter %d, old %u\n", tid, iteration, old);
            }
            if (old < MAX) {
                // Increment shared memory counter
                shared_count[threadIdx.x]++;
            }
        }

        // Write shared memory result to global memory
        d_count[tid] += shared_count[threadIdx.x];
    }
}