#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cube(float *d_out, float *d_in) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global index
    float num = d_in[id];
    d_out[id] = num * num * num;
}