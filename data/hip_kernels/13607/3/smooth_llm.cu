#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth(float *v_new, const float *v) {
    int totalThreads = blockDim.x * gridDim.x; // Pre-calculate total threads
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x; // Corrected thread index calculation

    // Early exit if index is out of bounds
    if (myIdx >= totalThreads) return; 
    
    int myLeftIdx = (myIdx == 0) ? 0 : myIdx - 1;
    int myRightIdx = (myIdx == totalThreads - 1) ? totalThreads - 1 : myIdx + 1;
    
    float myElt = v[myIdx];
    float myLeftElt = v[myLeftIdx];
    float myRightElt = v[myRightIdx];
    
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
}