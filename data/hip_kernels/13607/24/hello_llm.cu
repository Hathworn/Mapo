#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use a block-based message to improve performance
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello world! I'm thread %d\n", tid);
}