#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void remove_redness_from_coordinates( const unsigned int*  d_coordinates, unsigned char* d_r, unsigned char* d_b, unsigned char* d_g, unsigned char* d_r_output, int    num_coordinates, int    num_pixels_y, int    num_pixels_x, int    template_half_height, int    template_half_width )
{
    int ny = num_pixels_y;
    int nx = num_pixels_x;
    int global_index_1d = ( blockIdx.x * blockDim.x ) + threadIdx.x;

    int imgSize = num_pixels_x * num_pixels_y;

    if ( global_index_1d < num_coordinates )
    {
        unsigned int image_index_1d = d_coordinates[ imgSize - global_index_1d - 1 ];
        ushort2 image_index_2d = make_ushort2(image_index_1d % num_pixels_x, image_index_1d / num_pixels_x);

        // Loop over the template area with boundary check
        for ( int y = max(0, image_index_2d.y - template_half_height); y <= min(ny - 1, image_index_2d.y + template_half_height); y++ )
        {
            for ( int x = max(0, image_index_2d.x - template_half_width); x <= min(nx - 1, image_index_2d.x + template_half_width); x++ )
            {
                int image_offset_index_1d_clamped = (ny * y) + x;

                // Compute average and store directly
                d_r_output[ image_offset_index_1d_clamped ] = (d_g[ image_offset_index_1d_clamped ] + d_b[ image_offset_index_1d_clamped ]) / 2;
            }
        }
    }
}