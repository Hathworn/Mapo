#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void batcherBitonicMergesort64(float * d_out, const float * d_in)
{
    // you are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid  = threadIdx.x;
    // Load input into shared memory
    sdata[tid] = d_in[tid];
    __syncthreads();

    // Perform bitonic sort
    for (int stage = 0; stage <= 5; stage++)
    {
        for (int substage = stage; substage >= 0; substage--)
        {
            int pairDistance = 1 << substage;                // Calculate distance of pair
            int index = (2 * pairDistance * (tid / pairDistance)) + (tid % pairDistance); // Compute index of the element to compare with
            if (index < tid)
            {
                if ((tid & pairDistance) == 0)               // Compare and swap for ascending order
                {
                    if (sdata[tid] > sdata[tid + pairDistance])
                    {
                        float temp = sdata[tid];
                        sdata[tid] = sdata[tid + pairDistance];
                        sdata[tid + pairDistance] = temp;
                    }
                }
                else                                         // Compare and swap for descending order
                {
                    if (sdata[tid] < sdata[tid + pairDistance])
                    {
                        float temp = sdata[tid];
                        sdata[tid] = sdata[tid + pairDistance];
                        sdata[tid + pairDistance] = temp;
                    }
                }
            }
            __syncthreads();
        }
    }
    // Copy result to output
    d_out[tid] = sdata[tid];
}