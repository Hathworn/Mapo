#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    if (myId < BIN_COUNT)  // Ensure myId is within bounds
    {
        int myItem = d_in[myId];
        int myBin = myItem % BIN_COUNT;
        atomicAdd(&(d_bins[myBin]), 1);  // Use atomic operation for safe bin increment
    }
}