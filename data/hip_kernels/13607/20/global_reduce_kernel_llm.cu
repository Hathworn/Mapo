#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
    extern __shared__ float s_data[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // Load shared memory from global memory
    s_data[tid] = d_in[myId];
    __syncthreads();

    // Do reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            s_data[tid] += s_data[tid + s];
        }
        __syncthreads();  // Ensure all adds at one stage are done
    }

    // Only thread 0 writes result for this block back to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = s_data[0];
    }
}