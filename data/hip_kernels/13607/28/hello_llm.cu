#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hello()
{
    // Use block and thread indices to provide unique identification for each thread
    printf("Hello world! I'm a thread %d in block %d\n", threadIdx.x, blockIdx.x);
}