#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    // Calculate normalization constant once for the entire kernel
    const float normalization_constant = 1.f / d_input_cdf[n - 1];

    // Use grid-stride loop for better performance and more flexibility in the number of threads
    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int index = global_index_1d; index < n; index += stride) {
        unsigned int input_value = d_input_cdf[index];
        float output_value = input_value * normalization_constant;
        d_output_cdf[index] = output_value;
    }
}