#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
    // Load input value into shared memory
    s[threadIdx.x] = p;
    __syncthreads();

    // Use warp-level reduction
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        if (threadIdx.x < offset) {
            s[threadIdx.x] += s[threadIdx.x + offset];
        }
        __syncthreads();
    }
    return s[0];
}

__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in)
{
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    if (t == 0)
    {
        *d_out_shared = sr;
    }
}