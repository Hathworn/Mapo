#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shmem_reduce_kernel(float *d_out, const float *d_in)
{
    // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid = threadIdx.x;

    // load shared mem from global mem
    sdata[tid] = d_in[myId];
    __syncthreads(); // make sure entire block is loaded!

    // unroll the loop to do reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // make sure all adds at one stage are done!
    }

    // perform warp-level reduction for remaining elements
    if (tid < 32)
    {
        volatile float *vsdata = sdata; // avoid bank conflicts
        vsdata[tid] += vsdata[tid + 32];
        vsdata[tid] += vsdata[tid + 16];
        vsdata[tid] += vsdata[tid + 8];
        vsdata[tid] += vsdata[tid + 4];
        vsdata[tid] += vsdata[tid + 2];
        vsdata[tid] += vsdata[tid + 1];
    }

    // only thread 0 writes result for this block back to global mem
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}