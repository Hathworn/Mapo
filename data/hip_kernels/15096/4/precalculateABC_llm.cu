#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void precalculateABC(float4* ABCm, float* M, float timestep, float alpha, unsigned int numPoints)
{
    // Calculate unique thread index
    int me_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if index exceeds number of points
    if (me_idx >= numPoints)
        return;

    // Precompute constants for optimization
    float twodelta = timestep * 2.0f;
    float deltasqr = timestep * timestep;

    // Load mass
    float Mii = M[me_idx];
    float Dii = alpha * Mii;  // Apply mass-proportional damping

    // Compute coefficients using precomputed constants
    float Ai = 1.0f / (Dii / twodelta + Mii / deltasqr);
    float Bi = (2.0f * Mii / deltasqr) * Ai;
    float Ci = (Dii / twodelta) * Ai - 0.5f * Bi;

    // Store results in the output array
    ABCm[me_idx] = make_float4(Ai, Bi, Ci, Mii);
}