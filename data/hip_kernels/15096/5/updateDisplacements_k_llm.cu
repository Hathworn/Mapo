#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void updateDisplacements_k(float4 *Ui_t, float4 *Ui_tminusdt, float *M, float4 *Ri, float4 *Fi, int maxNumForces, float4 *ABC, unsigned int numPoints)
{
    int me_idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (me_idx >= numPoints)
        return;

    float4 F = make_float4(0, 0, 0, 0);

    // Unroll the loop for efficiency
    for (int i = 0; i < maxNumForces; i += 4)
    {
        float4 force1 = Fi[me_idx * maxNumForces + i];
        float4 force2 = (i + 1 < maxNumForces) ? Fi[me_idx * maxNumForces + i + 1] : make_float4(0, 0, 0, 0);
        float4 force3 = (i + 2 < maxNumForces) ? Fi[me_idx * maxNumForces + i + 2] : make_float4(0, 0, 0, 0);
        float4 force4 = (i + 3 < maxNumForces) ? Fi[me_idx * maxNumForces + i + 3] : make_float4(0, 0, 0, 0);

        F.x += (force1.x + force2.x + force3.x + force4.x);
        F.y += (force1.y + force2.y + force3.y + force4.y);
        F.z += (force1.z + force2.z + force3.z + force4.z);
    }

    float4 ABCi = ABC[me_idx];
    float4 Uit = Ui_t[me_idx];
    float4 Uitminusdt = Ui_tminusdt[me_idx];

    float4 R = Ri[me_idx];
    float x = ABCi.x * (R.x - F.x) + ABCi.y * Uit.x + ABCi.z * Uitminusdt.x;
    float y = ABCi.x * (R.y - F.y) + ABCi.y * Uit.y + ABCi.z * Uitminusdt.y;
    float z = ABCi.x * (R.z - F.z) + ABCi.y * Uit.z + ABCi.z * Uitminusdt.z;

    Ui_tminusdt[me_idx] = make_float4(x, y, z, 0);
}