#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Avoid redundant global memory access by using a register
    float acc = *accuracy;  
    acc /= N;  // Perform division in register
    *accuracy = acc;  // Write back result to global memory
}