#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_mult(int *a, int *b, int *c, int N) {
    // Calculate row and column index based on block and thread indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    // Ensure within bounds before computating
    if (col < N && row < N) {
        // Use shared memory to store matrices for faster access
        __shared__ int shared_a[16][16];
        __shared__ int shared_b[16][16];

        for (int i = 0; i < (N + 15) / 16; ++i) {
            // Load data into shared memory
            if (i * 16 + threadIdx.x < N && row < N)
                shared_a[threadIdx.y][threadIdx.x] = a[row * N + i * 16 + threadIdx.x];
            else
                shared_a[threadIdx.y][threadIdx.x] = 0;

            if (i * 16 + threadIdx.y < N && col < N)
                shared_b[threadIdx.y][threadIdx.x] = b[(i * 16 + threadIdx.y) * N + col];
            else
                shared_b[threadIdx.y][threadIdx.x] = 0;

            __syncthreads();

            // Calculate partial sum
            for (int j = 0; j < 16; ++j) {
                sum += shared_a[threadIdx.y][j] * shared_b[j][threadIdx.x];
            }
            __syncthreads();
        }
        // Store result in global memory
        c[row * N + col] = sum;
    }
}