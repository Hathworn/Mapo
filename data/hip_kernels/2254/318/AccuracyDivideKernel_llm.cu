#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use shared memory for faster access
    __shared__ float shared_accuracy;
    
    // Only one thread performs the division
    if (threadIdx.x == 0) {
        shared_accuracy = *accuracy / N;
    }
    
    // Synchronize threads to ensure the shared value is updated
    __syncthreads();

    // All threads write back the updated shared value
    *accuracy = shared_accuracy;
}