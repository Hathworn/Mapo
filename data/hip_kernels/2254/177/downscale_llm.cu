#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv( int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii/d3;
    z = ii % d2;
    ii = ii/d2;
    y = ii % d1;
    ii = ii/d1;
    x = ii;
    w = w*scale_factor+off_x;
    z = z*scale_factor+off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x*d1+y)*d2)+z)*d3+w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii/d3;
    z = ii % d2;
    ii = ii/d2;
    y = ii % d1;
    ii = ii/d1;
    x = ii;
    w = w/scale_factor;
    z = z/scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x*d1+y)*d2)+z)*d3+w;
}

__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    // Calculate unique thread index
    long ii = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * blockIdx.y * gridDim.y);

    // Ensure thread index is within array bounds
    if (ii >= no_elements) return;

    // Use shared memory to reduce memory latency
    extern __shared__ float shared_gradOutput[];

    // Load data into shared memory: Use a single loop to avoid redundancy
    for (int i = 0; i < scale_factor * scale_factor; i++) {
        int off_x = i % scale_factor;
        int off_y = i / scale_factor;
        int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, off_x, off_y);
        shared_gradOutput[i] = gradOutput_data[ipidx];
    }

    __syncthreads(); // Synchronize threads to ensure shared memory is populated

    // Sum up the contributions from the scaled positions
    for (int i = 0; i < scale_factor * scale_factor; i++) {
        gradInput_data[ii] += shared_gradOutput[i];
    }
}