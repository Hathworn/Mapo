#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multMatriz(float *da, float *db, float *dc, int num) {
    // Calculate global index for matrix element
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure within matrix bounds
    if(i < num && j < num) {
        float sum = 0.0f; // Initialize sum for dot product
        // Perform dot product for element (i, j)
        for (unsigned int k = 0; k < num; k++) {
            sum += da[i * num + k] * db[k * num + j];
        }
        dc[i * num + j] = sum; // Store result in matrix C
    }
}