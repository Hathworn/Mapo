#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void indices() {
    // Calculate unique thread ID within the grid
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    double id_sqrt = sqrt((double)id); // Compute square root once and save result

    // Optimized by removing unnecessary synchronization call
    printf("blockdimy: %d  threadx: %d  Blockidx: %d  blockdimx: %d id: %d raiz: %f\n",
           blockDim.y, threadIdx.x, blockIdx.x, blockDim.x, id, id_sqrt);
}