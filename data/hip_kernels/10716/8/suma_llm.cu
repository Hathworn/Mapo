#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void suma(int a, int b, int *c){
    // Use shared memory to store the result
    __shared__ int temp;
    if (threadIdx.x == 0) {
        temp = a + b;
    }
    __syncthreads();
    
    // Write result to global memory with a single thread
    if (threadIdx.x == 0) {
        *c = temp;
    }
}