#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpymm(double *y, double *a, double *b, int m, int n, int p) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    if (bid < m) {
        if (tid < n) {
            int offset = bid * n * p + tid * p;
            for (int c = 0; c < p; c++) {
                dots_s[offset + c] = a[bid * n + tid] * b[tid * p + c];
            }
        }

        __syncthreads();

        // Using reduction pattern for better parallelism.
        if (tid < n / 2) {
            for (int c = 0; c < p; c++) {
                for (int step = 1; step < n; step *= 2) {
                    if (tid + step < n) {
                        dots_s[bid * n * p + tid * p + c] += dots_s[bid * n * p + (tid + step) * p + c];
                    }
                    __syncthreads();
                }
            }
        }

        if (tid == 0) {
            for (int c = 0; c < p; c++) {
                y[bid * p + c] = dots_s[bid * n * p + c];
            }
        }
    }
}