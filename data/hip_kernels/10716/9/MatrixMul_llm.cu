#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMul( float *Md , float *Nd , float *Pd , const int WIDTH )
{
    // Calculate thread id
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;
    
    // Ensure position is within bounds
    if (row < WIDTH && col < WIDTH) {
        float Pvalue = 0.0f;
        // Optimize memory access by using local variable for result
        for (int k = 0; k < WIDTH; ++k) {
            Pvalue += Md[row * WIDTH + k] * Nd[k * WIDTH + col];
        }
        Pd[row * WIDTH + col] = Pvalue;
    }
}