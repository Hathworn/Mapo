#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multMatriz(float *da, float *db, float *dc, int num) {
    // Calculate global row and column indices
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    // Use local registers for sum and loop variable
    float sum = 0.0f;
    if (i < num && j < num) {
        for (unsigned int k = 0; k < num; k++) {
            sum += da[i * num + k] * db[k * num + j];
        }
        dc[i * num + j] = sum;
    }
}