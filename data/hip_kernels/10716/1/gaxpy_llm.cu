#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpy(double *y, double *a, double *x, int m, int n) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    if (bid < m) {
        // Cooperative loading of data into shared memory
        double local_sum = 0.0;
        for (int i = tid; i < n; i += blockDim.x) {
            local_sum += a[bid * n + i] * x[i];
        }

        // Reduce sum within each block
        dots_s[tid] = local_sum;
        __syncthreads();

        // Reduction step
        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
            if (tid < stride) {
                dots_s[tid] += dots_s[tid + stride];
            }
            __syncthreads();
        }

        // Write result back to global memory
        if (tid == 0) {
            y[bid] = dots_s[0];
        }
    }
}