#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulSh(float *Md, float *Nd, float *Pd, const int WIDTH) {
    // Use shared memory for tiles
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // Calculate thread coordinates
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

    // Initialize register for accumulation to reduce global memory accesses
    float Pvalue = 0;

    for (int m = 0; m < WIDTH / TILE_WIDTH; m++) {
        // Load Md and Nd tiles into shared memory
        Mds[threadIdx.y][threadIdx.x] = Md[row * WIDTH + (m * TILE_WIDTH + threadIdx.x)];
        Nds[threadIdx.y][threadIdx.x] = Nd[(m * TILE_WIDTH + threadIdx.y) * WIDTH + col];
        __syncthreads();  // Synchronize threads after loading

        // Perform computation on tiles
        for (int k = 0; k < TILE_WIDTH; k++) {
            Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        }
        __syncthreads();  // Synchronize threads before starting next tile phase
    }

    // Store result in global memory
    Pd[row * WIDTH + col] = Pvalue;
}