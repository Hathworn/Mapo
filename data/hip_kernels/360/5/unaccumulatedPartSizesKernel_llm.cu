#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unaccumulatedPartSizesKernel(int size, int *accumulatedSize, int *sizes) {
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if within bounds
    if (idx < size) {
        // Efficient memory access
        int prev_accumulated = (idx > 0) ? accumulatedSize[idx - 1] : 0;
        sizes[idx] = accumulatedSize[idx] - prev_accumulated;
    }
}