#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void makeSwapsKernel(int size, int *partition, int *partSizes, int *nodeWeights, int *swap_to, int *swap_from, int *swap_index, float *desirability)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (idx < size) 
    {
        // Simplifying condition checking
        bool valid_swap = (idx == size - 1) || (swap_from[idx] != swap_from[idx + 1]);
        
        if (desirability[idx] > 0.1 && valid_swap) 
        {
            int swapTo = swap_to[idx];
            int swapFrom = swap_from[idx];
            int swapIndex = swap_index[idx];
            int nodeWeight = nodeWeights[swapIndex];
            
            partition[swapIndex] = swapTo;
            atomicAdd(&partSizes[swapTo], nodeWeight);
            atomicAdd(&partSizes[swapFrom], -nodeWeight);
            
            // Debug print can be optionally enabled
            // printf("Swapping node: %d, %d from part: %d, %d to part: %d, %d desirability: %f\n", swapIndex, nodeWeight, swapFrom, partSizes[swapFrom], swapTo, partSizes[swapTo], desirability[idx]);
        }
    }
}