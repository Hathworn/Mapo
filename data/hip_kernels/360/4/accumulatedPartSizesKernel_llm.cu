#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulatedPartSizesKernel(int size, int *part, int *weights, int *accumulatedSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int thisPart = part[idx];
        // Using atomic add for safe concurrent updates
        if (idx == size - 1 || thisPart != part[idx + 1]) {
            atomicAdd(&accumulatedSize[thisPart], weights[idx]);
        }
    }
}