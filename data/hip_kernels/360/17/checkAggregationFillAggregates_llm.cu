#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void checkAggregationFillAggregates(int size, int *adjIndices, int *adjacency, int* aggregation, int* valuesIn, int* valuesOut, int* incomplete) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < size) {
        int currentVal = valuesIn[idx];
        int currentAgg = aggregation[idx];
        int start = adjIndices[idx];
        int end = adjIndices[idx + 1];

        // Use shared memory to cache data
        __shared__ int sharedValuesIn[256];

        // Load values into shared memory for faster access
        if (threadIdx.x < size) {
            sharedValuesIn[threadIdx.x] = valuesIn[threadIdx.x];
        }
        __syncthreads(); // Ensure shared memory is populated

        for (int i = start; i < end; i++) {
            int neighborIdx = adjacency[i];
            int neighborAgg = aggregation[neighborIdx];
            int neighborVal = sharedValuesIn[neighborIdx]; // Use shared memory

            if (neighborAgg == currentAgg && neighborVal > currentVal) {
                currentVal = neighborVal;
                atomicExch(incomplete, 1); // Use atomic operation for thread safety
            }
        }

        valuesOut[idx] = currentVal;
    }
}