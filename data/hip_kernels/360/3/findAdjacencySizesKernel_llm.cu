#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findAdjacencySizesKernel(int size, int *adjIndexes, int *output) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Load adjacent indices into shared memory for coalesced access
        __shared__ int sharedAdj[1024];
        sharedAdj[threadIdx.x] = adjIndexes[idx];
        __syncthreads();

        if (threadIdx.x < blockDim.x - 1)
            output[idx] = sharedAdj[threadIdx.x + 1] - sharedAdj[threadIdx.x];
        else if (idx + 1 < size)
            output[idx] = adjIndexes[idx + 1] - sharedAdj[threadIdx.x];
    }
}