#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mapAdjacencyToBlockKernel(int size, int *adjIndexes, int *adjacency, int *adjacencyBlockLabel, int *blockMappedAdjacency, int *fineAggregate) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int begin = adjIndexes[idx];
        int end = adjIndexes[idx + 1];
        int thisBlock = fineAggregate[idx];

        // Use shared memory to reduce global memory access
        __shared__ int sharedFineAggregate[256]; // Assuming max blockDim.x is 256
        if (threadIdx.x < 256)
            sharedFineAggregate[threadIdx.x] = fineAggregate[threadIdx.x];
        __syncthreads();

        // Fill block labeled adjacency and block mapped adjacency vectors
        for (int i = begin; i < end; i++) {
            int neighbor = sharedFineAggregate[adjacency[i]];

            if (thisBlock == neighbor) {
                adjacencyBlockLabel[i] = -1;
                blockMappedAdjacency[i] = -1;
            } else {
                adjacencyBlockLabel[i] = thisBlock;
                blockMappedAdjacency[i] = neighbor;
            }
        }
    }
}