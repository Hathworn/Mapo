#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Initialize(int size, unsigned int *randoms, int *bestSeen, int *origin, int *mis, int *incomplete) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure idx is within bounds before execution
    if (idx >= size) return;

    // Taus-step calculation optimized
    unsigned int z = randoms[idx];
    unsigned int b = (((z << 13) ^ z) >> 19);
    z = (((z & UINT_MAX) << 12) ^ b);

    // Set the origin to be self
    origin[idx] = idx;

    // Set the bestSeen value based on mis status
    bestSeen[idx] = (mis[idx] == 1) ? 1000001 : (mis[idx] == -1 ? (z % 1000000) : 0);

    // Write out new random value for seeding
    randoms[idx] = z;

    // Only reset incomplete if idx is 0
    if (idx == 0)
        incomplete[0] = 0;
}