#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MakeMerges(int size, int *mergeWith, int *offsets, int *mis) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int currentAgg = mis[idx];
        int newAgg = mergeWith[currentAgg];
        // Use conditional operator for concise conditional execution
        mis[idx] = (newAgg == -1) ? (currentAgg - offsets[currentAgg]) : (newAgg - offsets[newAgg]);
    }
}