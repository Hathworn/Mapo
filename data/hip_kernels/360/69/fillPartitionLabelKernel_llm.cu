#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fillPartitionLabelKernel(int size, int *coarseAggregate, int *fineAggregateSort, int *partitionLabel)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for faster access in the same block
    extern __shared__ int sharedCoarseAggregate[];

    if (threadIdx.x < size) {
        sharedCoarseAggregate[threadIdx.x] = coarseAggregate[threadIdx.x];
    }

    __syncthreads();

    if (idx < size) {
        partitionLabel[idx] = sharedCoarseAggregate[fineAggregateSort[idx]];
    }
}