#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getAggregateStartIndicesKernel(int size, int *fineAggregateSort, int *aggregateRemapIndex)
{
    // Calculate global thread index using block and thread indices.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure idx is within bounds and execute only when necessary.
    if (idx < size)
    {
        // Use shared memory to minimize global memory access latency.
        __shared__ int sharedFineAggregateSort[256]; // Assuming a block size of 256 for simplification.
        
        // Each thread loads its element into shared memory.
        sharedFineAggregateSort[threadIdx.x] = fineAggregateSort[idx];
        __syncthreads(); // Synchronize to ensure shared memory is fully populated.
       
        // Perform the boundary check and comparison within shared memory.
        if (idx == 0 || sharedFineAggregateSort[threadIdx.x] != sharedFineAggregateSort[threadIdx.x - 1])
        {
            // Write the result back to global memory.
            aggregateRemapIndex[sharedFineAggregateSort[threadIdx.x]] = idx;
        }
    }
}