#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MakeSplits_Large(int baseAggregateIdx, int* splitting, int* aggregation, int* aggMapAdjIndices, int* aggMapAdjacency, int* adjIndices, int* adjacency) {
    int currentAgg = splitting[blockIdx.x];
    int aggBegin = aggMapAdjIndices[currentAgg];
    int aggSize = aggMapAdjIndices[currentAgg + 1] - aggBegin;
    int newAgg = baseAggregateIdx + blockIdx.x;

    __shared__ int nodeIds[256];
    __shared__ int scratchA[256];
    __shared__ int scratchB[256];
    __shared__ int rootA;
    __shared__ int rootB;
    __shared__ int aCount, bCount;
    __shared__ bool incomplete;

    if (threadIdx.x < aggSize)
        nodeIds[threadIdx.x] = aggMapAdjacency[aggBegin + threadIdx.x];
    __syncthreads();

    // Declare outside if-loop to avoid multiple declarations
    int neighbors[40];
    int nextNeighbor = 0;
    int nodeId = -1;
    if (threadIdx.x < aggSize) {
        nodeId = nodeIds[threadIdx.x];
        int start = adjIndices[nodeId];
        int end = adjIndices[nodeId + 1];

        // Optimize the binary search loop
        for (int i = start; i < end; i++) {
            int neighborId = adjacency[i];
            int lo = 0;
            int hi = aggSize - 1;
            while (lo <= hi) {
                int mid = lo + (hi - lo) / 2;
                if (nodeIds[mid] < neighborId)
                    lo = mid + 1;
                else
                    hi = mid - 1;
            }
            if (lo < aggSize && nodeIds[lo] == neighborId) {
                neighbors[nextNeighbor++] = lo;
            }
        }
    }
    __syncthreads();

    // Initiate distance calculation
    if (threadIdx.x < aggSize) {
        scratchA[threadIdx.x] = (threadIdx.x == 0) ? 0 : -1;
    }

    int myDist = (threadIdx.x == 0) ? 0 : -1;
    bool swapped = false;
    incomplete = true;
    __syncthreads();

    // Optimize the while-loop
    while (incomplete) {
        incomplete = false;
        __syncthreads();

        if (threadIdx.x < aggSize && myDist == -1) {
            for (int i = 0; i < nextNeighbor; i++) {
                int neighborDist = scratchA[neighbors[i]];
                if (neighborDist > -1) myDist = neighborDist + 1;
            }
        }
        __syncthreads();

        if (threadIdx.x < aggSize && myDist > 0 && !swapped) {
            swapped = true;
            scratchA[threadIdx.x] = myDist;
            if (threadIdx.x == 0) rootA = threadIdx.x;
            incomplete = true;
        }
        __syncthreads();
    }

    if (threadIdx.x < aggSize) {
        scratchA[threadIdx.x] = (threadIdx.x == rootA) ? 0 : -1;
    }

    myDist = (threadIdx.x == rootA) ? 0 : -1;
    swapped = false;
    incomplete = true;
    __syncthreads();

    while (incomplete) {
        incomplete = false;
        __syncthreads();

        if (threadIdx.x < aggSize && myDist == -1) {
            for (int i = 0; i < nextNeighbor; i++) {
                int neighborDist = scratchA[neighbors[i]];
                if (neighborDist > -1) myDist = neighborDist + 1;
            }
        }
        __syncthreads();

        if (threadIdx.x < aggSize && myDist > 0 && !swapped) {
            swapped = true;
            scratchA[threadIdx.x] = myDist;
            if (threadIdx.x == rootA) rootB = threadIdx.x;
            incomplete = true;
        }
        __syncthreads();
    }

    int myAggregate = -1;
    if (threadIdx.x == rootA) myAggregate = currentAgg;
    if (threadIdx.x == rootB) myAggregate = newAgg;

    if (threadIdx.x < aggSize) {
        scratchA[threadIdx.x] = myAggregate;
        scratchB[threadIdx.x] = (myAggregate > -1) ? 0 : -1;
    }
    
    incomplete = true;
    __syncthreads();

    while (incomplete) {
        incomplete = false;
        __syncthreads();

        if (threadIdx.x < aggSize && myAggregate == -1) {
            for (int i = 0; i < nextNeighbor; i++) {
                int neighborAgg = scratchA[neighbors[i]];
                if (neighborAgg > -1) {
                    myDist = scratchB[neighbors[i]] + 1;
                    myAggregate = neighborAgg;
                }
            }
            if (myAggregate == -1) incomplete = true;
            if (myAggregate == newAgg) atomicAdd(&bCount, 1);
            if (myAggregate == currentAgg) atomicAdd(&aCount, 1);
        }
        __syncthreads();

        if (threadIdx.x < aggSize) {
            scratchA[threadIdx.x] = myAggregate;
            scratchB[threadIdx.x] = myDist;
        }
        __syncthreads();
    }

    int sizeDifference = abs(aCount - bCount);
    bool moveToA = aCount < bCount;
    __shared__ int moved;
    moved = 0;
    int toMove = sizeDifference / 2;
    incomplete = true;
    __syncthreads();

    while (incomplete && moved < toMove) {
        incomplete = false;
        __syncthreads();
        bool swapping = false;
        int newDist = INT_MAX;
        if (threadIdx.x < aggSize) {
            bool canSwap = (moveToA) ? (myAggregate == newAgg) : (myAggregate == currentAgg);
            bool borderNode = false;

            for (int i = 0; i < nextNeighbor; i++) {
                int neighborAgg = scratchA[neighbors[i]];
                int neighborDist = scratchB[neighbors[i]];
                if (neighborAgg == myAggregate && neighborDist > myDist) canSwap = false;

                if (neighborAgg != myAggregate) {
                    newDist = min(newDist, neighborDist + 1);
                    borderNode = true;
                }
            }

            if (borderNode && canSwap && atomicAdd(&moved, 1) < toMove) {
                swapping = true;
            }
        }
        __syncthreads();
        if (swapping) {
            int a = moveToA ? 1 : -1;
            atomicAdd(&bCount, -a);
            atomicAdd(&aCount, a);
            scratchA[threadIdx.x] = moveToA ? currentAgg : newAgg;
            scratchB[threadIdx.x] = newDist;
            incomplete = true;
        }
        __syncthreads();
    }

    if (threadIdx.x < aggSize) {
        aggregation[nodeId] = scratchA[threadIdx.x];
    }
}