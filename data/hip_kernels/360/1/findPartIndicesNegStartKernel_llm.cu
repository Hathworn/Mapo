#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findPartIndicesNegStartKernel(int size, int *array, int *partIndices) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if (idx < size) {
        int value = array[idx];
        int nextValue = array[idx + 1];
        
        // Check for changes in value and update partIndices using atomic operation
        if (value != nextValue) {
            atomicExch(&partIndices[value + 1], idx);
        }
    }
}