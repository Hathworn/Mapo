#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void allocateNodesKernel(int size, int *adjIndexes, int *adjacency, int *partIn, int *partOut, int *aggregated)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure idx is within the valid range
    if(idx < size && aggregated[idx] == 0)
    {
        int start = adjIndexes[idx];
        int end = adjIndexes[idx + 1];

        // Use shared memory for reduced latency
        __shared__ int candidates[10];
        __shared__ int candidateCounts[10];

        // Initialize shared memory for this thread
        for(int i = threadIdx.x; i < 10; i += blockDim.x)
        {
            candidates[i] = -1;
            candidateCounts[i] = 0;
        }
        __syncthreads(); // Ensure initialization is completed

        // Traverse neighbors to find potential aggregations
        for(int i = start; i < end; i++)
        {
            int candidate = partIn[adjacency[i]];
            if(candidate != -1)
            {
                for(int j = 0; j < 10 && candidate != -1; j++)
                {
                    if(candidates[j] == -1)
                    {
                        candidates[j] = candidate;
                        candidateCounts[j] = 1;
                        break;
                    }
                    else if(candidates[j] == candidate)
                    {
                        candidateCounts[j] += 1;
                        break;
                    }
                }
            }
        }

        // Find the most frequent candidate
        int addTo = candidates[0];
        int count = candidateCounts[0];
        for(int i = 1; i < 10; i++)
        {
            if(candidateCounts[i] > count)
            {
                count = candidateCounts[i];
                addTo = candidates[i];
            }
        }
        partOut[idx] = addTo;
        if(addTo != -1)
        {
            aggregated[idx] = 1;
        }
    }
}