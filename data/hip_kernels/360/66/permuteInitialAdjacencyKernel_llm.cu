#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void permuteInitialAdjacencyKernel(int size, int *adjIndexesIn, int *adjacencyIn, int *permutedAdjIndexesIn, int *permutedAdjacencyIn, int *ipermutation, int *fineAggregate)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx < size)
    {
        // Load indices once to reduce global memory accesses
        int permutedIdx = ipermutation[idx];
        int oldBegin = adjIndexesIn[permutedIdx];
        int oldEnd = adjIndexesIn[permutedIdx + 1];
        int newBegin = permutedAdjIndexesIn[idx];
        
        // Use a shared variable to minimize recalculation
        int runSize = oldEnd - oldBegin;

        // Copy and transform adjacency in a single loop without unnecessary load/store
        for(int i = 0; i < runSize; i++)
        {
            int adjIdx = adjacencyIn[oldBegin + i];
            permutedAdjacencyIn[newBegin + i] = fineAggregate[adjIdx];
        }
    }
}