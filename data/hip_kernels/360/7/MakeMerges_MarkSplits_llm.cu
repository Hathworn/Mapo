#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MakeMerges_MarkSplits(int size, int* mergeWith, int* offsets, int* mis, int* splitsToMake) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return; // Return early if out of bounds

    int currentAgg = mis[idx];
    int newAgg = mergeWith[currentAgg];

    if (newAgg == -1) {
        mis[idx] = currentAgg - offsets[currentAgg]; // Apply offset when not merging
    } else {
        int newId = newAgg - offsets[newAgg];
        mis[idx] = newId;
        atomicExch(&splitsToMake[newId], 1); // Ensure atomic write
    }
}