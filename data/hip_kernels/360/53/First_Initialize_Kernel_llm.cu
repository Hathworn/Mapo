#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void First_Initialize_Kernel(int size, unsigned int *randoms, int *bestSeen, int *origin)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a grid-stride loop to handle more elements efficiently
    for (int i = idx; i < size; i += blockDim.x * gridDim.x) 
    {
        // Set the origin to be self
        origin[i] = i;

        // Set the bestSeen value to be random
        bestSeen[i] = randoms[i] % 1000000;
    }
}