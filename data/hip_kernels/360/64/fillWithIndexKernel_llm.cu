#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fillWithIndexKernel(int size, int *array)
{
    // Use shared memory for faster access
    extern __shared__ int sharedArray[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx < size)
    {
        // Copy data to shared memory
        sharedArray[threadIdx.x] = idx; 
        
        // Sync threads before copying back to global memory
        __syncthreads();

        // Write back from shared memory
        array[idx] = sharedArray[threadIdx.x];
    }
}