#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void removeRuntyPartsKernel(int size, int *partition, int *removeStencil, int *subtractions)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        int currentNode = partition[idx];
        // Using a ternary operator for branchless optimization
        partition[idx] = (removeStencil[currentNode] == 1) ? -1 : partition[idx] - subtractions[currentNode];
    }
}