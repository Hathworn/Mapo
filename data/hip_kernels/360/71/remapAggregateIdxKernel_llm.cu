#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void remapAggregateIdxKernel(int size, int *fineAggregateSort, int *aggregateRemapId)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure idx does not exceed the allowed range
    if(idx < size)
    {
        // Perform the mapping operation
        fineAggregateSort[idx] = aggregateRemapId[fineAggregateSort[idx]];
    }
}