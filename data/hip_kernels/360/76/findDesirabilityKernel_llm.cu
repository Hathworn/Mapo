#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findDesirabilityKernel(int size, int optimalSize, int *adjIndexes, int *adjacency, int *partition, int *partSizes, int *nodeWeights, int *swap_to, int *swap_from, int *swap_index, float *desirability)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        int currentPart = partition[idx];
        int currentPartSize = partSizes[currentPart];
        int nodeSize = nodeWeights[idx];
        int selfAdjacency = 0;
        int addTo = -1;
        float bestDesirability = 0;

        // The currentWeightFactor is higher the farther the count is from average
        float currentWeightFactor = (float)abs(currentPartSize - optimalSize) / optimalSize;
        // The self improvement is a measure of how much better this partitions size will be if the node is gone.
        float selfImprovement = (abs(currentPartSize - optimalSize) - abs((currentPartSize - nodeSize) - optimalSize)) * currentWeightFactor;
        if(selfImprovement > 0)
        {
            int start = adjIndexes[idx];
            int end = adjIndexes[idx + 1];

            // Arrays to store info about neighboring aggregates
            // Optimize candidate arrays to use shared memory for improved access time
            __shared__ int candidates[10];
            __shared__ int candidateCounts[10];
            
            if (threadIdx.x < 10) {
                candidates[threadIdx.x] = -1;
                candidateCounts[threadIdx.x] = 0;
            }
            __syncthreads();

            // Going through the neighbors:
            for(int i = start; i < end; i++)
            {
                int candidate = partition[adjacency[i]];
                if(candidate == currentPart)
                    selfAdjacency++;
                else
                {
                    for(int j = 0; j < 10; j++)
                    {
                        // More efficient checking and updating through shared memory
                        if(candidate != -1 && candidates[j] == -1) {
                            atomicMax(&candidates[j], candidate);
                            atomicAdd(&candidateCounts[j], 1);
                            candidate = -1;
                        } else if(candidates[j] == candidate) {
                            atomicAdd(&candidateCounts[j], 1);
                            candidate = -1;
                        }
                    }
                }
            }

            // Finding the best possible swap:
            for(int i = 1; i < 10; i++)
            {
                if(candidates[i] != -1)
                {
                    int neighborPart = candidates[i];
                    int neighborPartSize = partSizes[neighborPart];
                    float neighborWeightFactor = (float)abs(neighborPartSize - optimalSize) / optimalSize;
                    float neighborImprovement = ((float)(abs(neighborPartSize - optimalSize) - abs((neighborPartSize + nodeSize) - optimalSize))) * neighborWeightFactor;
                    // Combine with self improvement to get net
                    neighborImprovement += selfImprovement;
                    // Multiply by adjacency factor
                    neighborImprovement *= (float)candidateCounts[i] / selfAdjacency;

                    if(neighborImprovement > bestDesirability)
                    {
                        addTo = neighborPart;
                        bestDesirability = neighborImprovement;
                    }
                }
            }
        }

        swap_from[idx] = currentPart;
        swap_index[idx] = idx;
        swap_to[idx] = addTo;
        desirability[idx] = bestDesirability;
    }
}