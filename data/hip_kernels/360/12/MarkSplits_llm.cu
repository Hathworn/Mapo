#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MarkSplits(int size, bool force, int minPartSize, int maxPartSize, int* partSizes, int* splitsToMake) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;  // Early exit for out-of-bound threads

    int currentSize = partSizes[idx];
    bool shouldSplit = currentSize > maxPartSize && (force || currentSize > minPartSize * 2);
    splitsToMake[idx] = static_cast<int>(shouldSplit);  // Use type casting for clarity
}