#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Iterate(int size, int *originIn, int *originOut, int *bestSeenIn, int *bestSeenOut, int *adjIndexes, int *adjacency) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds
    if (idx < size) {
        // Cache values to reduce repeated global memory access
        int bestSeen = bestSeenIn[idx];
        int origin = originIn[idx];

        // Proceed only if bestSeen value is less than the threshold
        if (bestSeen < 1000001) {
            // Fetch starting and ending adjacency indices for the current node
            int start = adjIndexes[idx];
            int end = adjIndexes[idx + 1];

            // Use shared variables to minimize access to global memory and reduce latency
            for (int i = start; i < end; i++) {
                int neighbor = adjacency[i];
                int challenger = bestSeenIn[neighbor];
                int challengerOrigin = originIn[neighbor];

                // Update origin if challenger has the same bestSeen value but higher origin
                if (challenger > 0 && challenger == bestSeen && challengerOrigin > origin) {
                    origin = challengerOrigin;
                }

                // Update bestSeen and origin if a better challenger is found
                if (challenger > bestSeen) {
                    bestSeen = challenger;
                    origin = challengerOrigin;
                }
            }
        }

        // Write updated values back to global memory
        bestSeenOut[idx] = bestSeen;
        originOut[idx] = origin;
    }
}