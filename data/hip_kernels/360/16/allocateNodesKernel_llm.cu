#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void allocateNodesKernel(int size, int *adjIndexes, int *adjacency, int *partIn, int *partOut, int *aggregated) {
    // Calculate the unique index for each thread.
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the index is within the size.
    if (idx < size) {
        // Proceed if the node is not aggregated yet.
        if (aggregated[idx] == 0) {
            int start = adjIndexes[idx];
            int end = adjIndexes[idx + 1];

            // Use shared memory for faster access and to reduce global memory reads.
            __shared__ int candidates[10];
            __shared__ int candidateCounts[10];

            // Initialize first 10 elements (threads 0-9).
            if (threadIdx.x < 10) {
                candidates[threadIdx.x] = -1;
                candidateCounts[threadIdx.x] = 0;
            }

            // Synchronize to make sure initialization is complete.
            __syncthreads();
            
            // Iterate over the node's neighbors
            for (int i = start; i < end; i++) {
                int candidate = partIn[adjacency[i]];
                if (candidate != -1) {
                    bool inserted = false;
                    
                    for (int j = 0; j < 10 && !inserted; j++) {
                        if (candidates[j] == candidate) {
                            atomicAdd(&candidateCounts[j], 1);  // Use atomic for safe increments.
                            inserted = true;
                        } else if (candidates[j] == -1) {
                            candidates[j] = candidate;
                            atomicAdd(&candidateCounts[j], 1);  // Use atomic for safe increments.
                            inserted = true;
                        }
                    }
                }
            }

            // More synchronizations to ensure all candidate counts have been updated.
            __syncthreads();

            // Determine the candidate with the most connections.
            if (threadIdx.x == 0) {
                int addTo = candidates[0];
                int count = candidateCounts[0];
                for (int i = 1; i < 10; i++) {
                    if (candidateCounts[i] > count) {
                        count = candidateCounts[i];
                        addTo = candidates[i];
                    }
                }
                
                // Assign the node to the most connected aggregate.
                partOut[idx] = addTo;
                if (addTo != -1) {
                    aggregated[idx] = 1;
                }
            }
        }
    }
}