#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PreInitialize(int size, unsigned int *randoms, int *bestSeen, int *origin, int *mis) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a loop stride pattern for increased parallel efficiency
    for (int i = idx; i < size; i += blockDim.x * gridDim.x) {
        // Set the origin to be self
        origin[i] = i;

        // Set MIS to be -1;
        mis[i] = -1;

        // Set the bestSeen value to be random
        bestSeen[i] = randoms[i] % 1000000;
    }
}