#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MarkMerges(int size, int* desiredMerges, int* merging, int* mergesToMake, int* incomplete) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int desiredMerge = desiredMerges[idx];

        // Check if merger is valid and reciprocal
        if (desiredMerge >= 0 && desiredMerges[desiredMerge] == idx) {
            // Use ternary operator for concise conditionals
            int mergeIndex = desiredMerge > idx ? desiredMerge : idx;
            int targetIndex = desiredMerge > idx ? idx : desiredMerge;

            mergesToMake[mergeIndex] = targetIndex;
            merging[idx] = 1;
            merging[desiredMerge] = 1;
        } else if (desiredMerge >= 0) {
            incomplete[0] = 1; // Mark incomplete for further checks
        }
    }
}