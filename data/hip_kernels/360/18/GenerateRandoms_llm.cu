#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GenerateRandoms(int size, int iterations, unsigned int *randoms, unsigned int *seeds) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return; // Exit early if idx is beyond size, reducing unnecessary calculations.

    unsigned int z = seeds[idx];
    int offset = idx;
    int step = 32768;

    for (int i = 0; i < iterations; i++) {
        unsigned int b = (((z << 13) ^ z) >> 19);
        z = (((z & UINT_MAX) << 12) ^ b);
        randoms[offset] = z;
        offset += step;
        if (offset >= size) break; // Exit loop when offset is beyond size to save computation.
    }
}