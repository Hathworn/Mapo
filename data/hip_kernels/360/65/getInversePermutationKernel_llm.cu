#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getInversePermutationKernel(int size, int *original, int *inverse)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        // Use atomic operation to prevent data races
        atomicExch(&inverse[original[idx]], idx);
    }
}