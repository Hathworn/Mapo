#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindDesirableMergeSplits(int size, int minSize, int maxSize, int desiredSize, int* adjIndices, int* adjacency, int* partSizes, int* desiredMerges, int* merging) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        // Skip active aggregates (already merged or unmergeable)
        if (merging[idx] != 1) {
            int currentSize = partSizes[idx];
            bool currentOutSized = currentSize < minSize || currentSize > maxSize;
            float bestDesirability = 0;
            int mostDesirable = -1;
            int start = adjIndices[idx];
            int end = adjIndices[idx + 1];

            for (int i = start; i < end; i++) {
                int neighborAgg = adjacency[i];

                // Consider only active neighbor aggregates
                if (merging[neighborAgg] != 1) {
                    int neighborSize = partSizes[neighborAgg];
                    bool neighborOutSized = neighborSize < minSize || neighborSize > maxSize;
                    int totalSize = currentSize + neighborSize;

                    // Calculate desirability for legal pairs
                    bool legalPair = (neighborOutSized || currentOutSized) && totalSize > minSize * 2 && totalSize < maxSize * 2;
                    float desirability = legalPair ? 1.0f / abs(desiredSize - totalSize) : 0;

                    // Update most desirable neighbor
                    if (desirability > bestDesirability) {
                        bestDesirability = desirability;
                        mostDesirable = neighborAgg;
                    }
                }
            }

            if (mostDesirable == -1) {
                merging[idx] = 1;
            }

            desiredMerges[idx] = mostDesirable;
        }
    }
}