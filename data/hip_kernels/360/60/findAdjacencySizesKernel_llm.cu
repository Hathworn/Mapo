#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findAdjacencySizesKernel(int size, int *adjIndexes, int *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure thread operates within valid range
    if (idx >= size) return;
    output[idx] = adjIndexes[idx + 1] - adjIndexes[idx];
}