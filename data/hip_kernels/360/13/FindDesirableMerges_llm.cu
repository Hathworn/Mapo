#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindDesirableMerges(int size, int minSize, int maxSize, bool force, int* adjIndices, int* adjacency, int* partSizes, int* desiredMerges, int* merging) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return; // Simplified boundary check

    if (merging[idx] == 1) return; // Skip active aggregates early

    // Cache current size to register
    int currentSize = partSizes[idx];
    int checkedNeighbors = 0;
    float bestDesirability = 0.0f;
    int mostDesirable = -1;
    int start = adjIndices[idx];
    int end = adjIndices[idx + 1];

    for (int i = start; i < end; i++) {
        int neighborAgg = adjacency[i];
        
        if (merging[neighborAgg] == 1) continue; // Skip active neighbors early

        checkedNeighbors++;
        int neighborSize = partSizes[neighborAgg];
        
        // Compute desirability with branchless operations
        float desirability = max(0.0f, minSize - currentSize) + max(0.0f, minSize - neighborSize);
        int totalSize = currentSize + neighborSize;
        desirability *= (totalSize > maxSize && !force) ? 0.0f : 1.0f / max(1.0f, totalSize - maxSize);

        // Update most desirable neighbor
        if (desirability > bestDesirability) {
            bestDesirability = desirability;
            mostDesirable = neighborAgg;
        }
    }

    if (mostDesirable == -1) merging[idx] = 1; // Mark aggregate as active if no desirable merges found

    // Log only if force is true and most desirable merge was not found
    if (currentSize < minSize && force && mostDesirable == -1)
        printf("Aggregate %d is too small but found no merges! %d / %d neighbors checked.\n", idx, checkedNeighbors, end - start);

    desiredMerges[idx] = mostDesirable; // Update desired merges
}