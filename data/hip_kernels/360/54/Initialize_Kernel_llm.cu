#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Initialize_Kernel(int size, unsigned int *randoms, int *bestSeen, int *origin, int *mis, int *incomplete)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
    {
        // Optimize random number generation using bit manipulation
        unsigned int z = randoms[idx];
        unsigned int b = (((z << 13) ^ z) >> 19);
        z = (((z & UINT_MAX) << 12) ^ b);

        // Initialize origin and bestSeen
        origin[idx] = idx;
        int status = mis[idx];
        bestSeen[idx] = (status == -1) ? (z % 1000000) : (status == 1 ? 1000001 : 0);

        // Update randoms with new generated value
        randoms[idx] = z;
    }

    // Efficiently reset incomplete value once, ensuring it only runs when idx is 0
    if (idx == 0)
    {
        incomplete[0] = 0;
    }
}