#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulatedPartSizesKernel(int size, int *part, int *weights, int *accumulatedSize)
{
    // Use shared memory to minimize global memory accesses.
    __shared__ int sharedWeights[256];
  
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
    // Load data into shared memory, if within bounds.
    if (idx < size) {
        sharedWeights[threadIdx.x] = weights[idx];
        __syncthreads();
        
        int thisPart = part[idx];
        
        // Update accumulatedSize using shared memory, if within bounds.
        if (idx == size - 1 || thisPart != part[idx + 1]) {
            atomicAdd(&accumulatedSize[thisPart], sharedWeights[threadIdx.x]);
        }
    }
}