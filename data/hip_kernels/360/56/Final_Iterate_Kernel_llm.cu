#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Final_Iterate_Kernel(int size, int *originIn, int *originOut, int *bestSeenIn, int *bestSeenOut, int *adjIndexes, int *adjacency, int *mis, int *incomplete)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return; // Early exit for out-of-bound threads

    int bestSeen = bestSeenIn[idx];
    int origin = originIn[idx];

    if (bestSeen < 1000001)
    {
        int start = adjIndexes[idx];
        int end = adjIndexes[idx + 1];

        // Unroll loop for improved performance
        #pragma unroll
        for (int i = start; i < end; i++)
        {
            int neighbor = adjacency[i];
            unsigned int challenger = bestSeenIn[neighbor];
            int challengerOrigin = originIn[neighbor];

            if (challenger > bestSeen || (challenger == bestSeen && challengerOrigin > origin))
            {
                bestSeen = challenger;
                origin = challengerOrigin;
            }
        }
    }

    // Write new MIS status
    int misStatus = (origin == idx) ? 1 : ((bestSeen == 1000001) ? 0 : -1);
    mis[idx] = misStatus;

    // Atomic OR to mark incomplete if still unassigned
    if (misStatus == -1)
    {
        atomicOr(incomplete, 1);
    }
}