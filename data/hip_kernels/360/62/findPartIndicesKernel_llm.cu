#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void findPartIndicesKernel(int size, int *array, int *partIndices)
{
    // Optimize memory accesses using shared memory
    extern __shared__ int sharedArray[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        // Load data into shared memory
        sharedArray[threadIdx.x] = array[idx];
        __syncthreads();

        int value = sharedArray[threadIdx.x];
        int nextValue = (idx != size - 1 && threadIdx.x < blockDim.x - 1) ? sharedArray[threadIdx.x + 1] : array[idx + 1];
        
        // Atomic operation to prevent race conditions
        if (value != nextValue)
        {
            atomicMax(&partIndices[value + 1], idx + 1);
        }
    }
}