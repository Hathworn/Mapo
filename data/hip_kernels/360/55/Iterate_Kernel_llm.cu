#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Iterate_Kernel(int size, int *originIn, int *originOut, int *bestSeenIn, int *bestSeenOut, int *adjIndexes, int *adjacency)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx >= size) return;  // Use a guard clause for better readability and efficiency

    int bestSeen = bestSeenIn[idx];
    int origin = originIn[idx];
    
    if (bestSeen >= 1000001) {
        bestSeenOut[idx] = bestSeen;  // Directly assign unchanged values where applicable.
        originOut[idx] = origin;
        return;
    }

    int start = adjIndexes[idx];
    int end = adjIndexes[idx + 1];

    // Optimized loop to minimize redundant conditional checks
    for (int i = start; i < end; i++)
    {
        int neighbor = adjacency[i];
        int challenger = bestSeenIn[neighbor];
        int challengerOrigin = originIn[neighbor];

        // Use logical operators to reduce branching
        if ((challenger > bestSeen) || (challenger == bestSeen && challengerOrigin > origin))
        {
            bestSeen = challenger;
            origin = challengerOrigin;
        }
    }

    // Write out the best values found
    bestSeenOut[idx] = bestSeen;
    originOut[idx] = origin;
}