#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Generate_Randoms_Kernel(int size, int iterations, unsigned int *randoms, unsigned int *seeds)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int z = seeds[idx];
    int offset = idx;
    const int step = 32768;

    // Optimize loop by removing conditional branch and adjusting loop count
    int maxOffset = size / step * step + idx;
    for (int i = 0; i < iterations && offset < maxOffset; i++)
    {
        unsigned int b = (((z << 13) ^ z) >> 19);
        z = (((z & UINT_MAX) << 12) ^ b);
        randoms[offset] = z;
        offset += step;
    }
}