#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findPartIndicesKernel(int size, int *array, int *partIndices) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size - 1) { // Prevents unnecessary checks for the last index
        int value = array[idx];
        int nextValue = array[idx + 1];
        if (value != nextValue) {
            partIndices[value + 1] = idx + 1;
        }
    } else if (idx == size - 1) { // Handles the boundary condition for the last index
        int value = array[idx];
        partIndices[value + 1] = idx + 1;
    }
}