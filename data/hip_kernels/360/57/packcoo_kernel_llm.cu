#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void packcoo_kernel(int num_entries, int* row_indices, int* column_indices, int* aggridx, int* partidx, int* partlabel)
{
    // Calculate global thread index
    int entryidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread is within bounds
    if(entryidx < num_entries)
    {
        int row = row_indices[entryidx];
        int col = column_indices[entryidx];
        int l = partlabel[row];

        // Cache partstart to avoid multiple memory accesses 
        int partstart = aggridx[partidx[l]];

        // Use bitwise operations efficiently
        unsigned int newindex = (row - partstart) << 16 | (col - partstart);

        // Store computed newindex back to row_indices
        row_indices[entryidx] = newindex;
    }
}