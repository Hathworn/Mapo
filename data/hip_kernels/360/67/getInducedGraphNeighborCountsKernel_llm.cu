#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void getInducedGraphNeighborCountsKernel(int size, int *aggregateIdx, int *adjIndexesOut, int *permutedAdjIndexes, int *permutedAdjacencyIn)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        int Begin = permutedAdjIndexes[aggregateIdx[idx]];
        int End = permutedAdjIndexes[aggregateIdx[idx + 1]];

        // Use shared memory for temporary sorting space
        extern __shared__ int sharedMem[];
        int* tempArray = &sharedMem[threadIdx.x * (End - Begin)];
        for(int i = Begin; i < End; i++)
        {
            tempArray[i - Begin] = permutedAdjacencyIn[i];
        }
        // Apply better sorting algorithm
        for (int i = 0; i < End - Begin - 1; ++i) {
            int minIdx = i;
            for (int j = i + 1; j < End - Begin; ++j) {
                if (tempArray[j] < tempArray[minIdx]) {
                    minIdx = j;
                }
            }
            if (minIdx != i) {
                int temp = tempArray[i];
                tempArray[i] = tempArray[minIdx];
                tempArray[minIdx] = temp;
            }
        }

        // Scan through the sorted adjacency to get the condensed adjacency
        int neighborCount = 0;
        for(int i = 0; i < End - Begin; i++)
        {
            if(tempArray[i] != idx && (i == 0 || tempArray[i] != tempArray[i - 1]))
            {
                permutedAdjacencyIn[neighborCount + Begin] = tempArray[i];
                neighborCount++;
            }
        }

        // Store the size
        adjIndexesOut[idx] = neighborCount;
    }
}
```
