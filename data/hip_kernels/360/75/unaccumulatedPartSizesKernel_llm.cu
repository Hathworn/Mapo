#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unaccumulatedPartSizesKernel(int size, int *accumulatedSize, int *sizes)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load accumulatedSize into shared memory for improved access speed
    extern __shared__ int sharedAccumulated[];
    
    if (idx < size) {
        sharedAccumulated[threadIdx.x] = accumulatedSize[idx];
        __syncthreads(); // Synchronize to ensure all threads have loaded their data

        // Calculate sizes using shared memory
        if (idx == 0) {
            sizes[idx] = sharedAccumulated[0];
        } else {
            sizes[idx] = sharedAccumulated[threadIdx.x] - sharedAccumulated[threadIdx.x - 1];
        }
    }
}