#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mapAdjacencyToBlockKernel(int size, int *adjIndexes, int *adjacency, int *adjacencyBlockLabel, int *blockMappedAdjacency, int *fineAggregate) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        int begin = adjIndexes[idx];
        int end = adjIndexes[idx + 1];
        int thisBlock = fineAggregate[idx];

        // Use shared memory for thread block wide reuse
        extern __shared__ int sharedMem[]; 
        int *sharedAdjacency = sharedMem;
        int *sharedFineAggregate = sharedMem + blockDim.x;

        // Load adjacency and fineAggregate data into shared memory
        if (threadIdx.x < end - begin) {
            sharedAdjacency[threadIdx.x] = adjacency[begin + threadIdx.x];
            sharedFineAggregate[threadIdx.x] = fineAggregate[sharedAdjacency[threadIdx.x]];
        }
        __syncthreads();

        // Fill block labeled adjacency and block mapped adjacency vectors
        for (int i = 0; i < end - begin; i++) {
            int neighbor = sharedFineAggregate[i];

            if (thisBlock == neighbor) { // Coalesce accesses to global memory
                adjacencyBlockLabel[begin + i] = -1;
                blockMappedAdjacency[begin + i] = -1;
            } else {
                adjacencyBlockLabel[begin + i] = thisBlock;
                blockMappedAdjacency[begin + i] = neighbor;
            }
        }
    }
}