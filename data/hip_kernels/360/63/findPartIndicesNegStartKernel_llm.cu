#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void findPartIndicesNegStartKernel(int size, int *array, int *partIndices)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1;
    if(idx < size)
    {
        int value = array[idx];
        int nextValue = array[idx + 1];
        // Reduce memory access by using shared memory
        __shared__ int sharedArray[256];
        int localIdx = threadIdx.x + 1;
        if(localIdx < blockDim.x)
        {
            sharedArray[localIdx] = array[idx];
            sharedArray[localIdx + 1] = array[idx + 1];
        }
        __syncthreads(); // Ensure all threads have loaded data into shared memory
        if(localIdx < blockDim.x && sharedArray[localIdx] != sharedArray[localIdx + 1])
        {
            partIndices[sharedArray[localIdx] + 1] = idx;
        }
    }
}