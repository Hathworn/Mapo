#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fillCondensedAdjacencyKernel(int size, int *aggregateIdx, int *adjIndexesOut, int *adjacencyOut, int *permutedAdjIndexesIn, int *permutedAdjacencyIn)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size)
    {
        int oldBegin = permutedAdjIndexesIn[ aggregateIdx[idx] ];
        int newBegin = adjIndexesOut[idx];
        int runSize = adjIndexesOut[idx + 1] - newBegin;

        // Use shared memory for temporary storage to optimize memory access
        extern __shared__ int sharedMem[];
        int* tempArray = sharedMem + threadIdx.x * runSize;

        // Copy adjacency over using shared memory
        for(int i = 0; i < runSize; i++)
        {
            tempArray[i] = permutedAdjacencyIn[oldBegin + i];
        }
        __syncthreads(); // Ensure all threads have completed the copy

        for(int i = 0; i < runSize; i++)
        {
            adjacencyOut[newBegin + i] = tempArray[i];
        }
    }
}