#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MakeSplits(int baseAggregateIdx, int* splitting, int* aggregation, int* aggMapAdjIndices, int* aggMapAdjacency, int* adjIndices, int* adjacency) {
    // Define shared memory sections
    __shared__ int nodeIds[64];
    __shared__ int scratchA[64];
    __shared__ int scratchB[64];
    __shared__ int rootA;
    __shared__ int rootB;
    __shared__ int aCount, bCount;
    __shared__ bool incomplete;
    __shared__ int moved;
    
    // Initialize shared variables
    incomplete = true;
    aCount = 1;
    bCount = 1;
    moved = 0;

    int currentAgg = splitting[blockIdx.x];
    int aggBegin = aggMapAdjIndices[currentAgg];
    int aggSize = aggMapAdjIndices[currentAgg + 1] - aggBegin;
    int newAgg = baseAggregateIdx + blockIdx.x;

    // Load in the node Id's from the aggregate map to the shared array:
    if (threadIdx.x < aggSize)
        nodeIds[threadIdx.x] = aggMapAdjacency[aggBegin + threadIdx.x];
    __syncthreads();

    // Each thread loads its neighbors list into registers
    int neighbors[40];
    int nextNeighbor = 0;
    int nodeId = -1;
    if (threadIdx.x < aggSize) {
        nodeId = nodeIds[threadIdx.x];
        int start = adjIndices[nodeId];
        int end = adjIndices[nodeId + 1];
        for (int i = start; i < end; i++) {
            int neighborId = adjacency[i];
            int a = 0, b = aggSize - 1, midpoint;
            while (a < b) {
                midpoint = a + ((b - a) / 2);
                if (nodeIds[midpoint] < neighborId)
                    a = midpoint + 1;
                else
                    b = midpoint;
            }

            if (nodeIds[a] == neighborId) {
                neighbors[nextNeighbor++] = a;
            }
        }
    }
    __syncthreads();

    // Begin distance calculation for first root
    if (threadIdx.x < aggSize)
        scratchA[threadIdx.x] = threadIdx.x == 0 ? 0 : -1;
    int myDist = threadIdx.x == 0 ? 0 : -1;
    bool swapped = false;
    
    while (incomplete) {
        incomplete = false;
        __syncthreads();

        if (threadIdx.x < aggSize && myDist == -1) {
            for (int i = 0; i < nextNeighbor; i++) {
                int neighborDist = scratchA[neighbors[i]];
                if (neighborDist > -1)
                    myDist = neighborDist + 1;
            }
        }
        __syncthreads();

        if (threadIdx.x < aggSize && myDist > 0 && !swapped) {
            swapped = true;
            scratchA[threadIdx.x] = myDist;
            rootA = threadIdx.x;
            incomplete = true;
        }
        __syncthreads();
    }

    // Begin distance calculation for second root
    if (threadIdx.x < aggSize)
        scratchA[threadIdx.x] = threadIdx.x == rootA ? 0 : -1;
    myDist = threadIdx.x == rootA ? 0 : -1;
    swapped = false;
    
    while (incomplete) {
        incomplete = false;
        __syncthreads();

        if (threadIdx.x < aggSize && myDist == -1) {
            for (int i = 0; i < nextNeighbor; i++) {
                int neighborDist = scratchA[neighbors[i]];
                if (neighborDist > -1) {
                    myDist = neighborDist + 1;
                }
            }
        }
        __syncthreads();

        if (threadIdx.x < aggSize && myDist > 0 && !swapped) {
            swapped = true;
            scratchA[threadIdx.x] = myDist;
            rootB = threadIdx.x;
            incomplete = true;
        }
        __syncthreads();
    }

    // Assign initial aggregates and distances
    int myAggregate = -1;
    if (threadIdx.x == rootA)
        myAggregate = currentAgg;
    if (threadIdx.x == rootB)
        myAggregate = newAgg;
    if (threadIdx.x < aggSize) {
        scratchA[threadIdx.x] = myAggregate;
        scratchB[threadIdx.x] = myAggregate > -1 ? 0 : -1;
    }
    incomplete = true;
    __syncthreads();

    // Assign nodes to aggregates
    while (incomplete) {
        incomplete = false;
        __syncthreads();

        if (threadIdx.x < aggSize && myAggregate == -1) {
            for (int i = 0; i < nextNeighbor; i++) {
                int neighborAgg = scratchA[neighbors[i]];
                if (neighborAgg > -1) {
                    myDist = scratchB[neighbors[i]] + 1;
                    myAggregate = neighborAgg;
                }
            }
            if (myAggregate == -1)
                incomplete = true;
            if (myAggregate == newAgg)
                atomicAdd((unsigned int*)&bCount, (unsigned)1);
            if (myAggregate == currentAgg)
                atomicAdd((unsigned int*)&aCount, (unsigned)1);
        }
        __syncthreads();

        if (threadIdx.x < aggSize) {
            scratchA[threadIdx.x] = myAggregate;
            scratchB[threadIdx.x] = myDist;
        }
        __syncthreads();
    }

    // Repair uneven split
    int sizeDifference = abs(aCount - bCount);
    bool moveToA = aCount < bCount;
    int toMove = sizeDifference / 2;

    while (incomplete && moved < toMove) {
        incomplete = false;
        __syncthreads();
        
        bool swapping = false;
        int newDist = INT_MAX;
        if (threadIdx.x < aggSize) {
            bool canSwap = moveToA ? myAggregate == newAgg : myAggregate == currentAgg;
            bool borderNode = false;

            for (int i = 0; i < nextNeighbor; i++) {
                int neighborAgg = scratchA[neighbors[i]];
                int neighborDist = scratchB[neighbors[i]];
                if (neighborAgg == myAggregate && neighborDist > myDist)
                    canSwap = false;

                if (neighborAgg != myAggregate) {
                    if (neighborDist + 1 < newDist)
                        newDist = neighborDist + 1;
                    borderNode = true;
                }
            }

            if (borderNode && canSwap && atomicAdd((unsigned int*)&moved, 1) < toMove) {
                swapping = true;
            }
        }
        __syncthreads();

        if (swapping) {
            int a = moveToA ? 1 : -1;
            atomicAdd((unsigned int*)&bCount, -a);
            atomicAdd((unsigned int*)&aCount, a);
            scratchA[threadIdx.x] = moveToA ? currentAgg : newAgg;
            scratchB[threadIdx.x] = newDist;
            incomplete = true;
        }
        __syncthreads();
    }

    // Write final aggregation results
    if (threadIdx.x < aggSize) {
        aggregation[nodeId] = scratchA[threadIdx.x];
    }
}