#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define PI 3.14159265359
#define DEG_TO_RAD (PI / 180.0)

typedef unsigned char byte;

__global__ void setup_kernel(hiprandState* state, unsigned long seed)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    // Use cooperative thread array indexing for better performance
    hiprandState localState;
    hiprand_init(seed, id, 0, &localState);
    state[id] = localState;
}