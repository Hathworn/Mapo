#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixAB(const int32_t* row_sum, const int32_t* col_sum, int32_t* output, int32_t K_A_B, int32_t N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (idx < N) { // Check if within array bounds
        output[blockIdx.x * N + idx] = K_A_B - row_sum[blockIdx.x] - col_sum[idx]; // Compute offset
    }
}