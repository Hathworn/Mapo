#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    // Cache row_sum[blockIdx.x] in a shared variable to reduce global memory access
    __shared__ int32_t cached_row_sum;
    if (threadIdx.x == 0) {
        cached_row_sum = row_sum[blockIdx.x];
    }
    __syncthreads();

    // Use the cached value for the entire block
    for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
        output[blockIdx.x * N + i] = -cached_row_sum;
    }
}