#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    hiprand_init(seed, idx, 0, &state[idx]); // Use unique index for each state
}