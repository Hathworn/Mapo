#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to compute offset of matrix A
__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Calculate global index
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds
    if (idx < N) {
        // Perform offset computation
        output[blockIdx.x * N + idx] = -col_sum[idx];
    }
}