#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    // Use shared memory to minimize redundant global memory access
    extern __shared__ int s_original[];
    int tid = threadIdx.x;

    // Calculate global index
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load data into shared memory
    if (global_idx < num_elements) {
        s_original[tid] = original[global_idx];
    }
    __syncthreads();

    for (; global_idx < num_elements; global_idx += blockDim.x * gridDim.x) {
        // Use shared memory value if within bound, otherwise reload from global memory
        int original_value = (global_idx < num_elements) ? s_original[tid] : original[global_idx];
        to_normalize[global_idx * 3] = batch_index;
        to_normalize[global_idx * 3 + 1] = class_index;
        to_normalize[global_idx * 3 + 2] = static_cast<int64_t>(original_value);

        // Update global index for next loop iteration
        global_idx += blockDim.x * gridDim.x;

        // Implicit barrier: sync threads before next shared memory access
        __syncthreads();

        if (global_idx < num_elements) {
            s_original[tid] = original[global_idx];
        }
        __syncthreads();
    }
}