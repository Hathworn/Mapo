#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    // Use a more efficient block index calculation
    int idx = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;
    if (idx < N)
    {
        float val = a[idx]; // Load value once
        a[idx] = val * val; // Square the value
    }
}