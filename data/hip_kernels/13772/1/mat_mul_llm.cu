#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define M  256
#define P  128
#define N   64
#define BLKSIZ 16

__global__ void mat_mul(float *Ad, float *Bd, float *Cd) {
    // Get global row and column indexes
    int Row = blockIdx.x * blockDim.x + threadIdx.x;
    int Col = blockIdx.y * blockDim.y + threadIdx.y;

    float c = 0.0f;

    // Shared memory for tiles of A and B
    __shared__ float As[BLKSIZ][BLKSIZ];
    __shared__ float Bs[BLKSIZ][BLKSIZ];

    // Iterate over tiles
    for (int t = 0; t < (P + BLKSIZ - 1) / BLKSIZ; ++t) {
        // Load tiles into shared memory
        if (Row < M && t * BLKSIZ + threadIdx.y < P) {
            As[threadIdx.x][threadIdx.y] = Ad[Row * P + t * BLKSIZ + threadIdx.y];
        } else {
            As[threadIdx.x][threadIdx.y] = 0.0f;
        }
        
        if (Col < N && t * BLKSIZ + threadIdx.x < P) {
            Bs[threadIdx.x][threadIdx.y] = Bd[(t * BLKSIZ + threadIdx.x) * N + Col];
        } else {
            Bs[threadIdx.x][threadIdx.y] = 0.0f;
        }

        __syncthreads();

        // Perform the computation
        for (int k = 0; k < BLKSIZ; ++k) {
            c += As[threadIdx.x][k] * Bs[k][threadIdx.y];
        }
        __syncthreads();
    }

    // Write the result
    if (Row < M && Col < N) {
        Cd[Row * N + Col] = c;
    }
}