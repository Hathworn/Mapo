#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rowMin(float* input, int* output, size_t rowS, size_t rowNum) {
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id < rowNum) {
        float temp[MAX_K / 2][2];
        size_t inId = id * rowS;

        // Load input into shared memory for coalesced global memory access
        __shared__ float sharedInput[MAX_K];
        for (int i = threadIdx.x; i < rowS; i += blockDim.x) {
            sharedInput[i] = input[inId + i];
        }
        __syncthreads();

        // Initialize temp array from shared memory
        for (int i = 0; i < rowS; i++) {
            temp[i][0] = sharedInput[i];
            temp[i][1] = (float)i;
        }

        // Optimize the selection sort with parallel reduction pattern if applicable
        for (int i = 0; i < rowS; i++) {
            float best = temp[i][0];
            int bestInd = i;
            for (int j = i + 1; j < rowS; j++) {  // Start from i+1
                if (temp[j][0] < best) {  // Corrected comparison direction for finding min
                    best = temp[j][0];
                    bestInd = j;
                }
            }
            // Swap only if a better minimum is found
            if (bestInd != i) {
                float iVal = temp[i][0];
                float iInd = temp[i][1];
                temp[i][0] = temp[bestInd][0];
                temp[i][1] = temp[bestInd][1];
                temp[bestInd][0] = iVal;
                temp[bestInd][1] = iInd;
            }
        }

        // Write the results back to global memory
        for (int i = 0; i < rowS; i++) {
            output[inId + i] = (int)temp[i][1];
        }
    }
}