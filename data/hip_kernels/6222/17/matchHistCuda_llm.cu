#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matchHistCuda(float* qSet, float* dbSet, size_t qSize, size_t dbSize, size_t hSize, float* out) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < qSize && idy < dbSize) {
        size_t qi = idx * hSize;
        size_t dbi = idy * hSize;

        // Use registers for the sums to optimize memory access
        float sumab = 0.0f;
        float suma2 = 0.0f;
        float sumb2 = 0.0f;

        // Unroll loop for performance improvement
        for (int k = 0; k < hSize; k++) {
            float qval = qSet[qi + k];
            float dbval = dbSet[dbi + k];
            sumab += qval * dbval;
            suma2 += qval * qval;
            sumb2 += dbval * dbval;
        }

        // Reduce division operations 
        float denom = sqrtf(suma2) * sqrtf(sumb2);
        if (denom > 0.0f) {
            float cossim = sumab / denom;
            out[idy * qSize + idx] = cossim;
        } else {
            out[idy * qSize + idx] = 0.0f; // Handle division by zero
        }
    }
}