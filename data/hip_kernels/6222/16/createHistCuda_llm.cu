#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void createHistCuda(float* siftCentroids, float* siftImage, int linesCent, int linesIm, float* temp)
{
    __shared__ float cosines[BLOCK_SIZE][2];

    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idy = blockIdx.y;
    size_t tid = threadIdx.x;

    if (idx < linesCent) {
        int centin = idx * 128;
        int imin = idy * 128;

        // Optimize cosine similarity calculation
        float sumab = 0;
        float suma2 = 0;
        float sumb2 = 0;

        #pragma unroll  // Use loop unrolling for better performance
        for (int k = 0; k < 128; k++) {
            float centVal = siftCentroids[centin + k];
            float imgVal = siftImage[imin + k];
            sumab += centVal * imgVal;
            suma2 += imgVal * imgVal;
            sumb2 += centVal * centVal;
        }

        float cossim = sumab / (sqrtf(suma2) * sqrtf(sumb2));  // Simplify division

        cosines[tid][0] = cossim;
        cosines[tid][1] = idx;

        __syncthreads();

        // Optimize for reduction to find the maximum cosine similarity
        for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (tid < s) {
                if (cosines[tid + s][0] > cosines[tid][0]) {
                    cosines[tid][0] = cosines[tid + s][0];
                    cosines[tid][1] = cosines[tid + s][1];
                }
            }
            __syncthreads();
        }

        if (tid == 0) {
            temp[(blockIdx.y * gridDim.x + blockIdx.x) * 2] = cosines[0][0];
            temp[(blockIdx.y * gridDim.x + blockIdx.x) * 2 + 1] = cosines[0][1];
        }
    }
}