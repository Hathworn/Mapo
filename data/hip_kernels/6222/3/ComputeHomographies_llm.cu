#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

//================= Device matching functions =====================//

template <int size>
__device__ void InvertMatrix(float elem[size][size], float res[size][size])
{
    int indx[size];
    float b[size];
    float vv[size];
    for (int i = 0; i < size; i++)
        indx[i] = 0;
    int imax = 0;
    float d = 1.0;
    for (int i = 0; i < size; i++) { // Find biggest element for each row
        float big = 0.0;
        for (int j = 0; j < size; j++) {
            float temp = fabs(elem[i][j]);
            if (temp > big)
                big = temp;
        }
        vv[i] = (big > 0.0) ? 1.0 / big : 1e16;
    }
    for (int j = 0; j < size; j++) {
        for (int i = 0; i < j; i++) {
            float sum = elem[i][j];
            for (int k = 0; k < i; k++)
                sum -= elem[i][k] * elem[k][j];
            elem[i][j] = sum;
        }
        float big = 0.0;
        for (int i = j; i < size; i++) {
            float sum = elem[i][j];
            for (int k = 0; k < j; k++)
                sum -= elem[i][k] * elem[k][j];
            elem[i][j] = sum;
            float dum = vv[i] * fabs(sum);
            if (dum >= big) {
                big = dum;
                imax = i;
            }
        }
        if (j != imax) {
            for (int k = 0; k < size; k++) {
                float dum = elem[imax][k];
                elem[imax][k] = elem[j][k];
                elem[j][k] = dum;
            }
            d = -d;
            vv[imax] = vv[j];
        }
        indx[j] = imax;
        if (elem[j][j] == 0.0)
            elem[j][j] = 1e-16;
        if (j != (size - 1)) {
            float dum = 1.0 / elem[j][j];
            for (int i = j + 1; i < size; i++)
                elem[i][j] *= dum;
        }
    }
    for (int j = 0; j < size; j++) {
        for (int k = 0; k < size; k++)
            b[k] = 0.0;
        b[j] = 1.0;
        int ii = -1;
        for (int i = 0; i < size; i++) {
            int ip = indx[i];
            float sum = b[ip];
            b[ip] = b[i];
            if (ii != -1)
                for (int j = ii; j < i; j++)
                    sum -= elem[i][j] * b[j];
            else if (sum != 0.0)
                ii = i;
            b[i] = sum;
        }
        for (int i = size - 1; i >= 0; i--) {
            float sum = b[i];
            for (int j = i + 1; j < size; j++)
                sum -= elem[i][j] * b[j];
            b[i] = sum / elem[i][i];
        }
        for (int i = 0; i < size; i++)
            res[i][j] = b[i];
    }
}

__global__ void ComputeHomographies(float *coord, int *randPts, float *homo, int numPts)
{
    float a[8][8], ia[8][8];
    float b[8];
    const int bx = blockIdx.x;
    const int tx = threadIdx.x;
    const int idx = blockDim.x * bx + tx;
    const int numLoops = blockDim.x * gridDim.x;

    #pragma unroll // Unroll the loop for performance
    for (int i = 0; i < 4; i++) {
        int pt = randPts[i * numLoops + idx];
        float x1 = coord[pt + 0 * numPts];
        float y1 = coord[pt + 1 * numPts];
        float x2 = coord[pt + 2 * numPts];
        float y2 = coord[pt + 3 * numPts];
        float *row1 = a[2 * i + 0];
        row1[0] = x1;
        row1[1] = y1;
        row1[2] = 1.0;
        row1[3] = row1[4] = row1[5] = 0.0;
        row1[6] = -x2 * x1;
        row1[7] = -x2 * y1;
        float *row2 = a[2 * i + 1];
        row2[0] = row2[1] = row2[2] = 0.0;
        row2[3] = x1;
        row2[4] = y1;
        row2[5] = 1.0;
        row2[6] = -y2 * x1;
        row2[7] = -y2 * y1;
        b[2 * i + 0] = x2;
        b[2 * i + 1] = y2;
    }

    InvertMatrix<8>(a, ia);
    __syncthreads(); // Ensure all threads reach this point

    #pragma unroll // Unroll the loop for performance
    for (int j = 0; j < 8; j++) {
        float sum = 0.0f;
        #pragma unroll // Unroll the loop for performance
        for (int i = 0; i < 8; i++)
            sum += ia[j][i] * b[i];
        homo[j * numLoops + idx] = sum;
    }
    __syncthreads(); // Ensure all threads reach this point
}