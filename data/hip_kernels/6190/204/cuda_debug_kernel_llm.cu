#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_debug_kernel()
{
    // Calculate the global thread index based on the 3D grid and block indices
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    const int idz = blockIdx.z * blockDim.z + threadIdx.z;

    // Ensure we're not accessing an out-of-bounds index
    if (idx >= gridDim.x * blockDim.x || 
        idy >= gridDim.y * blockDim.y ||
        idz >= gridDim.z * blockDim.z) {
        return;
    }

    // Add additional computation or data manipulation here if needed
}