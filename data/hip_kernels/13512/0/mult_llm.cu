#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
Copyright (C) 2009-2012 Fraunhofer SCAI, Schloss Birlinghoven, 53754 Sankt Augustin, Germany;
all rights reserved unless otherwise stated.

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful, but
WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place, Suite 330, Boston,
MA 02111-1307 USA
*/

/** Index function to address the two-dimensional arrays
Q and R

Matrices are stored in column-major order (like Fortran).

i is the row, j is the column (index starts at 1)
ld is the number of elements for each column
*/

#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

/* ---------------------------------------------------------------------- */

/*  Tuning can be done by different block sizes. */

#define BLOCK1 64

// 8800 GT:    128 x 1
// C1060:      128 x 1
#define BLOCK1X    64
#define BLOCK1Y    8

// 8800 GT:      64 x 4
// C1060:        64 x 8
#define BLOCK2X   512
#define BLOCK2Y   1

/** This kernel updates the matrix Q by a product of two vectors.

Q(1:m,k+1:n) -= R(k,k+1:n) * Q(1:m,k)

same as this BLAS-2 call:

call sger(M, N-K, -1.0, Q(1,K), 1, R(K,K+1), N, Q(1,K+1), M)

Each thread (t1,t2) will be responsible for BLOCK2X columns and BLOCK2Y
rows of the matrix Q.
*/

/**  QR factorization of a matrix

@param[in]      m is number of rows for Q and R
@param[in]      n is number of columns for Q and R
@param[in,out]  Q is a matrix of size m x n, column major order
@param[out]     R is a matrix of size m x n, column major order

@returns 0 if successful

Q(in) = Q(out) * R, where Q(out) is orthonormal and R upper-triangular
*/

__global__ void mult(float* Q, float* R, int m, int n, int k)
{
    __shared__ float RS[BLOCK1Y][BLOCK1X];
    __shared__ float QK[BLOCK1Y];

    int tid1 = threadIdx.x;
    int tid2 = threadIdx.y;

    int i = blockIdx.x * BLOCK1Y + tid2 + k;

    // Return early if out of bounds
    if (i < k || i > n) return;

    // Initialize local accumulators
    float S = 0.0f;

    // Use coalesced memory access
    for (int j = tid1 + 1; j <= m; j += BLOCK1X) {
        // Load QK once per row
        if (tid1 == 0) QK[tid2] = Q[IDX2F(j, k, m)];
        __syncthreads();
        S += QK[tid2] * Q[IDX2F(j, i, m)];
    }

    // Store in shared memory
    RS[tid2][tid1] = S;

    // Reduce in shared memory
    for (int NT = BLOCK1X >> 1; NT > 0; NT >>= 1) {
        __syncthreads();
        if (tid1 < NT) {
            RS[tid2][tid1] += RS[tid2][tid1 + NT];
        }
    }

    // Save result in R
    if (tid1 == 0) {
        R[IDX2F(k, i, n)] = RS[tid2][0];
    }
}