#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

#define BLOCK1 64
#define BLOCK1X 64
#define BLOCK1Y 8
#define BLOCK2X 512
#define BLOCK2Y 1

__global__ void scaleR(float* Q, float* R, int m, int n, int k, float S)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + k;

    // Check within valid column range
    if (i <= n) {
        R[IDX2F(k, i, n)] *= S;  // Scale element
    }
}