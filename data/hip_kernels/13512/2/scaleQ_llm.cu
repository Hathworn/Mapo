#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*
Copyright (C) 2009-2012 Fraunhofer SCAI, Schloss Birlinghoven, 53754 Sankt Augustin, Germany;
all rights reserved unless otherwise stated.

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation; either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful, but
WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 59 Temple Place, Suite 330, Boston,
MA 02111-1307 USA
*/

/** Index function to address the two-dimensional arrays
Q and R

Matrices are stored in column-major order (like Fortran).

i is the row, j is the column (index starts at 1)
ld is the number of elements for each column
*/

#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

/* ---------------------------------------------------------------------- */

/*  Tuning can be done by different block sizes. */

#define BLOCK1 64

// 8800 GT:    128 x 1
// C1060:      128 x 1
#define BLOCK1X    64
#define BLOCK1Y    8

// 8800 GT:      64 x 4
// C1060:        64 x 8
#define BLOCK2X   512
#define BLOCK2Y   1

/* ---------------------------------------------------------------------- */

/* Kernel for scaling the column k of the matrix Q. */

__global__ void scaleQ(float* Q, float* R, int m, int n, int k, float S)
{
    // Compute global row index for current thread
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    // Check for boundary condition
    if (row < m) {
        // Perform scaling operation
        Q[IDX2F(row + 1, k, m)] *= S;
    }
}