#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define IDX2F(i,j,ld) ((((j)-1)*(ld))+((i)-1))

#define BLOCK1 64
#define BLOCK1X    64
#define BLOCK1Y    8
#define BLOCK2X   512
#define BLOCK2Y   1

__global__ void update(float* Q, float* R, int m, int n, int k)
{
    __shared__ float RK[BLOCK2Y];
    __shared__ float QK[BLOCK2X];

    int tid1 = threadIdx.x;
    int tid2 = threadIdx.y;

    int j = blockIdx.y * BLOCK2Y + tid2 + k + 1;

    if (j < k + 1 || j > n) return;

    if (tid1 == 0) {
        RK[tid2] = R[IDX2F(k, j, n)];
    }

    // Improve data reuse by loading QK outside the loop
    if (tid2 == 0 && tid1 < m) {
        QK[tid1] = Q[IDX2F(tid1 + 1, k, m)];
    }

    __syncthreads();

    // Unroll loop for better performance
    for (int i = tid1 + 1; i <= m; i += BLOCK2X) {
        Q[IDX2F(i, j, m)] -= QK[tid1] * RK[tid2];
    }
}