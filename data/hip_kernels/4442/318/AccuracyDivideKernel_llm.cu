#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function with direct division
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use blockIdx.x to ensure only one division operation is performed
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        *accuracy /= N;
    }
}