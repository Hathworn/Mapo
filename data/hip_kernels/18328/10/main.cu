#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "GPU_simple_power_and_interbin_kernel.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float2 *d_input_complex = NULL;
hipMalloc(&d_input_complex, XSIZE*YSIZE);
float *d_output_power = NULL;
hipMalloc(&d_output_power, XSIZE*YSIZE);
int nTimesamples = 1;
float norm = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
GPU_simple_power_and_interbin_kernel<<<gridBlock, threadBlock>>>(d_input_complex,d_output_power,nTimesamples,norm);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
GPU_simple_power_and_interbin_kernel<<<gridBlock, threadBlock>>>(d_input_complex,d_output_power,nTimesamples,norm);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
GPU_simple_power_and_interbin_kernel<<<gridBlock, threadBlock>>>(d_input_complex,d_output_power,nTimesamples,norm);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}