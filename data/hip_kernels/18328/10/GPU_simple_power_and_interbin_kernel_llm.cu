#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GPU_simple_power_and_interbin_kernel(float2 *d_input_complex, float *d_output_power, int nTimesamples, float norm) {
    int pos_x = blockIdx.x * blockDim.x + threadIdx.x;
    int pos_y = blockIdx.y * nTimesamples;

    // Use shared memory for better access efficiency
    __shared__ float2 shared_data[MAX_THREADS_PER_BLOCK];

    // Load data into shared memory
    if (pos_x < nTimesamples) {
        shared_data[threadIdx.x] = d_input_complex[pos_y + pos_x];
    }
    __syncthreads();

    // Perform computations if within bounds
    if (pos_x < nTimesamples) {
        float2 A = shared_data[threadIdx.x];
        d_output_power[pos_y + pos_x] = (A.x * A.x + A.y * A.y) * norm;
    }
}
```
