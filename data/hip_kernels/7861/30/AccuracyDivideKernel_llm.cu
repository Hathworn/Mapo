#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Store the initial value of accuracy in a register
    float val = *accuracy;
    
    // Perform division in the register to reduce memory operations 
    val /= N;
    
    // Write the result back to global memory
    *accuracy = val;
}