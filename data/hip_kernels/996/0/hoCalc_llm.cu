#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hoCalc(double* rn, double* soilHeat, double* ho, int width_band) {
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (col < width_band) {  // Optimization: Check boundary condition once
        ho[col] = rn[col] - soilHeat[col];
    }
    // Removed unnecessary loop to improve performance and reduce memory access
}