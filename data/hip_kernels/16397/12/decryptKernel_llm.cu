#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void decryptKernel(char* deviceDataIn, char* deviceDataOut, int n) {
    // Calculate global index
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

    // Conditional execution if index is within bounds
    if (index < n) {
        // Perform data operation
        deviceDataOut[index] = deviceDataIn[index] - 1;
    }
}