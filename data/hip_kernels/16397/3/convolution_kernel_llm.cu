#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution_kernel(float *output, float *input, float *filter) {
    extern __shared__ float sh_input[];  // Use dynamic shared memory

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x * blockDim.x;
    int by = blockIdx.y * blockDim.y;

    int inputIndex = (by + ty) * inputWidth + (bx + tx);

    // Load input into shared memory
    sh_input[ty * blockDim.x + tx] = input[inputIndex];
   
    __syncthreads();  // Ensure all loads are visible to all threads in the block

    // Apply filter
    float result = 0.0f;
    for (int i = 0; i < filterHeight; ++i) {
        for (int j = 0; j < filterWidth; ++j) {
            int sharedIndex = (ty + i) * blockDim.x + (tx + j);
            result += sh_input[sharedIndex] * filter[i * filterWidth + j];
        }
    }

    output[inputIndex] = result;  // Store result in global memory
}