#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void encryptKernel(char* deviceDataIn, char* deviceDataOut, int n) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce global memory accesses
    __shared__ char sharedData[blockDim.x];
  
    if (index < n) {
        sharedData[threadIdx.x] = deviceDataIn[index];
        __syncthreads();
        deviceDataOut[index] = sharedData[threadIdx.x] + 1;
    }
}