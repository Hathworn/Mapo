#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vec_add_kernel(float *c, float *a, float *b, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    if (i < n) {  // Ensure index is within bounds
        c[i] = a[i] + b[i];  // Perform vector addition
    }
}