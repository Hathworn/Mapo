#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void vectorTransformKernel(float* A, float* B, float* Result) {
    // Calculate global index efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a more efficient boundary check
    if (i < 1000000) {
        // Avoid redundant memory access by using register for Result[i]
        float temp = A[i] * B[i];
        Result[i] += temp;
    }
}