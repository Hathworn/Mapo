#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAddKernel(float* A, float* B, float* Result) {
    // Optimize kernel with bounds check
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < gridDim.x * blockDim.x) {
        Result[i] = A[i] + B[i];
    }
}