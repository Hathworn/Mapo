#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAddKernel(float* A, float* B, float* Result) {
    // Calculate global index for the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Improve memory coalescing by loading inputs to registers
    float regA = A[i];
    float regB = B[i];

    // Perform the subtraction and store result in register before writing to global memory
    Result[i] = regA - regB;
}