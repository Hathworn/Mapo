#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAddKernel(float* __restrict__ A, float* __restrict__ B, float* __restrict__ Result) {
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform vector addition
    if (i < N) { // Assuming N is the total number of elements, consider boundary check
        Result[i] = __fadd_rn(A[i], B[i]); // Use faster intrinsic function for addition
    }
}