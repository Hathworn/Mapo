#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void decryptKernel(char* deviceDataIn, char* deviceDataOut, int n, char *key, int keySize) {
    // Calculate thread index
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;

    // Process only valid indices
    if (index < n) {
        // Optimized access using variable for repeated computation
        char keyChar = key[index % keySize];
        deviceDataOut[index] = deviceDataIn[index] - keyChar;
    }
}