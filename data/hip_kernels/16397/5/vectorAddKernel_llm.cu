#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAddKernel(float* A, float* B, float* Result) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data from global memory to registers (if needed for extra optimization)
    float a = A[i];
    float b = B[i];
    
    // Perform addition
    float result = a + b;
    
    // Write result back to global memory
    Result[i] = result;
}