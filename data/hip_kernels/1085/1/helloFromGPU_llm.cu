#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU(void)
{
    // Use faster branch-less logic with arithmetic
    const int selectedThread = 5; 
    printf("Hello World from GPU thread %d!\n", threadIdx.x * (threadIdx.x == selectedThread));
}