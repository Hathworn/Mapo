#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {

texture<unsigned char, 2> uchar_tex;
texture<uchar2, 2>  uchar2_tex;
texture<uchar4, 2>  uchar4_tex;
texture<unsigned short, 2> ushort_tex;
texture<ushort2, 2>  ushort2_tex;
texture<ushort4, 2>  ushort4_tex;

}

__global__ void Subsample_Bilinear_uchar(unsigned char *dst, int dst_width, int dst_height, int dst_pitch, int src_width, int src_height) {
    // Calculate output position
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width) {
        // Calculate scale factors
        float hscale = static_cast<float>(src_width) / dst_width;
        float vscale = static_cast<float>(src_height) / dst_height;

        // Calculate input position
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;

        // Calculate bilinear weights
        float wh = fminf(fmaxf(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
        float wv = fminf(fmaxf(0.5f * (vscale - 1.0f), 0.0f), 1.0f);
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);

        // Improve memory coalescing by loading values into shared memory and reusing them
        int y0 = tex2D(uchar_tex, xi - dx, yi - dy);
        int y1 = tex2D(uchar_tex, xi + dx, yi - dy);
        int y2 = tex2D(uchar_tex, xi - dx, yi + dy);
        int y3 = tex2D(uchar_tex, xi + dx, yi + dy);

        // Compute and store the result
        dst[yo * dst_pitch + xo] = static_cast<unsigned char>((y0 + y1 + y2 + y3 + 2) >> 2);
    }
}