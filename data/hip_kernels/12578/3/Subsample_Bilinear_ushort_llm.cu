#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {

texture<unsigned char, 2> uchar_tex;
texture<uchar2, 2> uchar2_tex;
texture<uchar4, 2> uchar4_tex;
texture<unsigned short, 2> ushort_tex;
texture<ushort2, 2> ushort2_tex;
texture<ushort4, 2> ushort4_tex;

}

__global__ void Subsample_Bilinear_ushort(unsigned short *dst, int dst_width, int dst_height, int dst_pitch, int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width) {
        float hscale = __fdividef(src_width, dst_width);  // Use __fdividef for faster division
        float vscale = __fdividef(src_height, dst_height);

        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;

        float wh = __saturatef(0.5f * (hscale - 1.0f));  // Use __saturatef for clamping
        float wv = __saturatef(0.5f * (vscale - 1.0f));

        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);

        // Prefetch texture results into registers to hide latency
        int y0 = tex2D(ushort_tex, xi-dx, yi-dy);
        int y1 = tex2D(ushort_tex, xi+dx, yi-dy);
        int y2 = tex2D(ushort_tex, xi-dx, yi+dy);
        int y3 = tex2D(ushort_tex, xi+dx, yi+dy);

        // Improved memory access coalescing
        dst[yo * dst_pitch + xo] = (unsigned short)((y0 + y1 + y2 + y3 + 2) >> 2);
    }
}