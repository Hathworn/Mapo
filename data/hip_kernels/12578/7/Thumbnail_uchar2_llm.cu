#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {

texture<unsigned char, 2> uchar_tex;
texture<uchar2, 2>  uchar2_tex;
texture<unsigned short, 2> ushort_tex;
texture<ushort2, 2>  ushort2_tex;
}

__global__ void Thumbnail_uchar2(int *histogram, int src_width, int src_height)
{
    // Calculate global thread positions
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure within bounds
    if (y < src_height && x < src_width)
    {
        // Fetch pixel from texture
        uchar2 pixel = tex2D(uchar2_tex, x, y);
        
        // Use shared memory to optimize atomic operations
        __shared__ int local_histogram[512];
        if (threadIdx.x < 512) local_histogram[threadIdx.x] = 0;
        __syncthreads();
        
        atomicAdd(&local_histogram[pixel.x], 1);
        atomicAdd(&local_histogram[256 + pixel.y], 1);
        __syncthreads();
        
        // Merge local histogram back into global histogram
        if (threadIdx.x < 512) atomicAdd(&histogram[threadIdx.x], local_histogram[threadIdx.x]);
    }
}