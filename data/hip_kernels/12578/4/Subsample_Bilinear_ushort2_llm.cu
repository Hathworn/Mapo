#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {

texture<unsigned char, 2> uchar_tex;
texture<uchar2, 2>  uchar2_tex;
texture<uchar4, 2>  uchar4_tex;
texture<unsigned short, 2> ushort_tex;
texture<ushort2, 2>  ushort2_tex;
texture<ushort4, 2>  ushort4_tex;

}

__global__ void Subsample_Bilinear_ushort2(ushort2 *dst, int dst_width, int dst_height, int dst_pitch2, int src_width, int src_height)
{
    int xo = blockIdx.x * blockDim.x + threadIdx.x;
    int yo = blockIdx.y * blockDim.y + threadIdx.y;

    if (yo < dst_height && xo < dst_width)
    {
        // Pre-compute scales
        float hscale = __fdividef((float)src_width, (float)dst_width);
        float vscale = __fdividef((float)src_height, (float)dst_height);
        float xi = (xo + 0.5f) * hscale;
        float yi = (yo + 0.5f) * vscale;

        // Use intrinsic functions for efficiency
        float wh = fminf(fmaxf(0.5f * (hscale - 1.0f), 0.0f), 1.0f);
        float wv = fminf(fmaxf(0.5f * (vscale - 1.0f), 0.0f), 1.0f);

        // Convert weights to two bilinear weights
        float dx = wh / (0.5f + wh);
        float dy = wv / (0.5f + wv);

        // Load texels and compute the bilinear filtered value
        ushort2 c0 = tex2D(ushort2_tex, xi-dx, yi-dy);
        ushort2 c1 = tex2D(ushort2_tex, xi+dx, yi-dy);
        ushort2 c2 = tex2D(ushort2_tex, xi-dx, yi+dy);
        ushort2 c3 = tex2D(ushort2_tex, xi+dx, yi+dy);

        // Accumulate results using efficient integer arithmetic
        int2 uv;
        uv.x = (static_cast<int>(c0.x) + static_cast<int>(c1.x) + static_cast<int>(c2.x) + static_cast<int>(c3.x) + 2) >> 2;
        uv.y = (static_cast<int>(c0.y) + static_cast<int>(c1.y) + static_cast<int>(c2.y) + static_cast<int>(c3.y) + 2) >> 2;

        // Store result
        dst[yo * dst_pitch2 + xo] = make_ushort2(static_cast<unsigned short>(uv.x), static_cast<unsigned short>(uv.y));
    }
}