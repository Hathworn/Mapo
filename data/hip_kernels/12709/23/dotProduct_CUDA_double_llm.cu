#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dotProduct_CUDA_double(double *sum, int size, double *vector1, double *vector2){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Sequential thread index across the blocks

    // Use shared memory to leverage memory coalescing
    __shared__ double sharedSum[256]; // 256 is exemplary, should match blockDim.x

    double temp = 0.0;
    if (idx < size) {
        temp = (vector2[idx]) * (vector1[idx]);
    }
    sharedSum[threadIdx.x] = temp;

    __syncthreads();

    // Perform reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            sharedSum[threadIdx.x] += sharedSum[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (threadIdx.x == 0) {
        atomicAdd(&sum[0], sharedSum[0]);
    }
}