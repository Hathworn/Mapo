#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_monte_carlo_d(double *estimate, hiprandState *states) {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int points_in_circle = 0;
    double x, y;

    // Initialize CURAND only once at the beginning
    if (threadIdx.x == 0) {
        hiprand_init(1234, blockIdx.x, 0, &states[blockIdx.x * blockDim.x]);
    }
    __syncthreads();
    hiprandState localState = states[tid]; // Cache state locally to reduce global memory access

    for (int i = 0; i < TRIALS_PER_THREAD; ++i) {
        x = hiprand_uniform(&localState);
        y = hiprand_uniform(&localState);
        points_in_circle += (x * x + y * y <= 1.0f); // count if (x, y) is in the circle
    }
    states[tid] = localState; // Store back the updated state
    estimate[tid] = 4.0f * points_in_circle / (double)TRIALS_PER_THREAD; // return estimate of pi
}