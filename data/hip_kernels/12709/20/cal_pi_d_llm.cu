#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cal_pi_d(double *sum, int nbin, double step, int nthreads, int nBLOCKS) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    double x;
    double sum_local = 0.0; // Use a local variable to reduce global memory access

    for (int i=idx; i < nbin; i += nthreads * nBLOCKS) {
        x = (i + 0.5) * step;
        sum_local += 4.0 / (1.0 + x * x);
    }

    sum[idx] = sum_local; // Store the result back to global memory once
}