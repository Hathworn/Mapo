#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_monte_carlo(float *estimate, hiprandState *states) {
    // Calculate unique thread index
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int points_in_circle = 0;
    float x, y;

    // Initialize CURAND with unique seed per thread
    hiprand_init(1234, tid, 0, &states[tid]);

    // Loop unrolling for performance improvement
    #pragma unroll
    for(int i = 0; i < TRIALS_PER_THREAD; i++) {
        x = hiprand_uniform(&states[tid]);
        y = hiprand_uniform(&states[tid]);
        points_in_circle += (x * x + y * y <= 1.0f); // count if x & y is in the circle.
    }

    // Efficiently calculate estimate of pi
    estimate[tid] = 4.0f * points_in_circle / (float) TRIALS_PER_THREAD;
}