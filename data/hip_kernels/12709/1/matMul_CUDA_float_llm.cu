#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NUM_THREAD  256  // Number of thread blocks
#define print(x) printf("%d",x)

float *matrixMul_float_serial(float vector1[], float vector2[], int size);
float *matrixMul_float_parallel(float vector1[], float vector2[], int size, int thread_count);
float *matrixMul_float_cuda(float* vector1, float* vector2, int num);
double *matrixMul_double_serial(double vector1[], double vector2[], int size);
double *matrixMul_double_parallel(double vector1[], double vector2[], int size, int thread_count);
double *matrixMul_double_cuda(double* vector1, double* vector2, int num);
double doubleGen();
float floatGen();
void operations(int size, int parallel, int serial, int cuda, int verify, int thread_count);
void print_results_float( int size, double time_spent);
void print_results_double( int size, double time_spent);
double verifyVectord(double *vector1, double *vector2, int size);
float verifyVectorf(float *vector1, float *vector2, int size);

__global__ void matMul_CUDA_float(float *sum, int size, float *vector1, float *vector2) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row = idx / size;  // Calculate row index
    int col = idx % size;  // Calculate column index

    // Ensure idx is within bounds
    if (idx < size * size) {
        float localSum = 0; // Use local variable for accumulation
        for (int k = 0; k < size; k++) {
            localSum += vector1[row * size + k] * vector2[k * size + col];
        }
        sum[idx] = localSum; // Write back to global memory once
    }
}