#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cal_pi(float *sum, int nbin, float step, int nthreads, int nBLOCKS) {
    // Use shared memory for reduction
    extern __shared__ float shared_sum[];
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Sequential thread index across the BLOCKS
    int tid = threadIdx.x;
    float x, local_sum = 0.0f;
    
    // Loop with striding across all blocks and threads
    for (int i = idx; i < nbin; i += nthreads * nBLOCKS) { 
        x = (i + 0.5f) * step;
        local_sum += 4.0f / (1.0f + x * x);
    }
    
    // Store local sum in shared memory
    shared_sum[tid] = local_sum;
    __syncthreads();
    
    // Reduce within block using shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_sum[tid] += shared_sum[tid + s];
        }
        __syncthreads();
    }
    
    // Write the result of the block sum to global memory
    if (tid == 0) {
        sum[blockIdx.x] = shared_sum[0];
    }
}