#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cal_pi(float *sum, int nbin, float step, int nthreads, int nblocks) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Sequential thread index across the blocks
    float local_sum = 0.0f; // Use a register to store the partial sum

    // Loop over the assigned work items
    for (int i = idx; i < nbin; i += nthreads * nblocks) {
        float x = (i + 0.5) * step; // Calculate x for this iteration
        local_sum += 4.0f / (1.0f + x * x); // Accumulate the partial sum
    }
    
    sum[idx] = local_sum; // Write the partial sum back to global memory
}