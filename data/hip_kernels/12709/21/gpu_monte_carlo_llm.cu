#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_monte_carlo(float *estimate, hiprandState *states) {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int points_in_circle = 0;
    float x, y;

    hiprandState localState = states[tid]; // Cache state in local memory for faster access
    hiprand_init(1234, tid, 0, &localState); // Initialize CURAND

    // Unroll the loop to improve performance
    for(int i = 0; i < TRIALS_PER_THREAD; i += 4) {
        x = hiprand_uniform(&localState);
        y = hiprand_uniform(&localState);
        points_in_circle += (x*x + y*y <= 1.0f);

        x = hiprand_uniform(&localState);
        y = hiprand_uniform(&localState);
        points_in_circle += (x*x + y*y <= 1.0f);

        x = hiprand_uniform(&localState);
        y = hiprand_uniform(&localState);
        points_in_circle += (x*x + y*y <= 1.0f);

        x = hiprand_uniform(&localState);
        y = hiprand_uniform(&localState);
        points_in_circle += (x*x + y*y <= 1.0f);
    }
    
    states[tid] = localState; // Store updated state back to global memory
    estimate[tid] = 4.0f * points_in_circle / (float) TRIALS_PER_THREAD; // return estimate of pi
}