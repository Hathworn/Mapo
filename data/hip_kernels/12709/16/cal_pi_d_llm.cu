#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cal_pi_d(double *sum, int nbin, double step, int nthreads, int nblocks) {

    extern __shared__ double sdata[];  // Shared memory for block-level reduction
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Sequential thread index across the blocks
    int tid = threadIdx.x;  // Local thread index within the block
    double x, mySum = 0.0;

    // Perform calculation and store intermediate results in mySum
    for (int i = idx; i < nbin; i += nthreads * nblocks) {
        x = (i + 0.5) * step;
        mySum += 4.0 / (1.0 + x * x);
    }

    // Store the result in shared memory for block reduction
    sdata[tid] = mySum;
    __syncthreads(); 

    // Perform block-level reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the block's result to the global sum
    if (tid == 0) {
        sum[blockIdx.x] = sdata[0];
    }
}