#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_monte_carlo_d(double *estimate, hiprandState *states) {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int points_in_circle = 0;
    double x, y;

    // Use shared memory for better performance
    __shared__ hiprandState localStates[BLOCK_SIZE];
    if (threadIdx.x < BLOCK_SIZE) {
        hiprand_init(1234, tid, 0, &localStates[threadIdx.x]);
    }
    __syncthreads();

    hiprandState localState = localStates[threadIdx.x];
    
    for(int i = 0; i < TRIALS_PER_THREAD; i++) {
        x = hiprand_uniform(&localState);
        y = hiprand_uniform(&localState);
        points_in_circle += (x * x + y * y <= 1.0f); // count if x & y is in the circle
    }

    localStates[threadIdx.x] = localState;
    __syncthreads();

    estimate[tid] = 4.0f * points_in_circle / (double) TRIALS_PER_THREAD; // return estimate of pi
}