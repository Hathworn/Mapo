#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_monte_carlo(float *estimate, hiprandState *states) {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int points_in_circle = 0;
    float x, y;

    // Initialize CURAND only once outside the loop to save time
    hiprand_init(1234, tid, 0, &states[tid]);

    // Use local variable register to cache state
    hiprandState localState = states[tid];

    for(int i = 0; i < TRIALS_PER_THREAD; i++) {
        // Update state with hiprand_uniform to increase efficiency
        x = hiprand_uniform(&localState);
        y = hiprand_uniform(&localState);
        points_in_circle += (x*x + y*y <= 1.0f); // count if x & y is in the circle.
    }

    // Write back to global memory
    states[tid] = localState;

    // Store the result
    estimate[tid] = 4.0f * points_in_circle / (float) TRIALS_PER_THREAD; // return estimate of pi
}