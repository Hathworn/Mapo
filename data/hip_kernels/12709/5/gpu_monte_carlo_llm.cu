#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_monte_carlo(float *estimate, hiprandState *states) {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int points_in_circle = 0;
    float x, y;

    hiprand_init(1234, tid, 0, &states[tid]); // Initialize CURAND

    // Unroll loop to optimize performance
    #pragma unroll 
    for(int i = 0; i < TRIALS_PER_THREAD; i++) {
        x = hiprand_uniform(&states[tid]);
        y = hiprand_uniform(&states[tid]);
        points_in_circle += (x * x + y * y <= 1.0f); // Count if x & y are in the circle
    }

    estimate[tid] = 4.0f * points_in_circle / (float) TRIALS_PER_THREAD; // Return estimate of pi
}