#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cal_pi(float *sum, int nbin, float step, int nthreads, int nblocks) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Sequential thread index across the blocks
    // Use shared memory for partial sums
    __shared__ float partial_sum[256];  // Assumes blockDim.x <= 256
    partial_sum[threadIdx.x] = 0.0f;

    for (int i = idx; i < nbin; i += nthreads * nblocks) {
        float x = (i + 0.5f) * step;
        partial_sum[threadIdx.x] += 4.0f / (1.0f + x * x);
    }
    
    // Synchronize threads in the block to ensure partial sums are written
    __syncthreads();
    
    // Reduce partial sums from all threads in the block
    if (threadIdx.x == 0) {
        float block_sum = 0.0f;
        for (int i = 0; i < blockDim.x; i++) {
            block_sum += partial_sum[i];
        }
        atomicAdd(sum, block_sum);  // Atomic add to global sum
    }
}