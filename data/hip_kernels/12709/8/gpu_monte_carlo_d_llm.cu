#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_monte_carlo_d(double *estimate, hiprandState *states) {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int points_in_circle = 0;
    double x, y;

    // Use thread index as seed for diversity
    hiprand_init(1234 + tid, tid, 0, &states[tid]);

    // Unroll loop for better parallelism
    #pragma unroll
    for(int i = 0; i < TRIALS_PER_THREAD; i++) {
        x = hiprand_uniform(&states[tid]);
        y = hiprand_uniform(&states[tid]);
        points_in_circle += (x * x + y * y <= 1.0f); // Check if point is within the circle
    }
    // Store computed estimate for Pi
    estimate[tid] = 4.0 * points_in_circle / TRIALS_PER_THREAD;
}