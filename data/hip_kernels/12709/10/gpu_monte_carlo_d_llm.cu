#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gpu_monte_carlo_d(double *estimate, hiprandState *states) {
    // Calculate global thread index
    unsigned int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int points_in_circle = 0; 
    double x, y;

    // Initialize CURAND
    hiprand_init(1234, tid, 0, &states[tid]);

    // Perform Monte Carlo simulation
    for (int i = 0; i < TRIALS_PER_THREAD; i++) {
        x = hiprand_uniform(&states[tid]);
        y = hiprand_uniform(&states[tid]);
        points_in_circle += (x * x + y * y <= 1.0f); // Check if the point is inside the circle
    }

    // Compute and store the estimate of pi
    estimate[tid] = 4.0 * points_in_circle / (double)TRIALS_PER_THREAD;
}