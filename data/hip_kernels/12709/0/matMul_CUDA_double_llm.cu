#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define NUM_THREAD  256
#define print(x) printf("%d",x)

float *matrixMul_float_serial(float vector1[], float vector2[], int size);
float *matrixMul_float_parallel(float vector1[], float vector2[], int size, int thread_count);
float *matrixMul_float_cuda(float* vector1, float* vector2, int num);
double *matrixMul_double_serial(double vector1[], double vector2[], int size);
double *matrixMul_double_parallel(double vector1[], double vector2[], int size, int thread_count);
double *matrixMul_double_cuda(double* vector1, double* vector2, int num);
double doubleGen();
float floatGen();
void operations(int size, int parallel, int serial, int cuda, int verify, int thread_count);
void print_results_float( int size, double time_spent);
void print_results_double( int size, double time_spent);
double verifyVectord(double *vector1, double *vector2, int size);
float verifyVectorf(float *vector1, float *vector2, int size);

__global__ void matMul_CUDA_double(double *sum, int size, double *vector1, double *vector2) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;  // Calculate row index
    int col = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate column index

    if (row < size && col < size) {
        double tempSum = 0.0;  // Local accumulator
        for (int k = 0; k < size; ++k) {
            tempSum += vector1[row * size + k] * vector2[k * size + col];  // Compute partial product
        }
        sum[row * size + col] = tempSum;  // Write result to global memory
    }
}