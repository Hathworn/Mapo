#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dotProduct_CUDA_float(float *sum, int size, float *vector1, float *vector2) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Sequential thread index across the blocks
    // Use shared memory to optimize access for parallel threads
    __shared__ float temp[1024];
    if (idx < size) {
        temp[threadIdx.x] = vector2[idx] * vector1[idx];
    } else {
        temp[threadIdx.x] = 0.0f;
    }
    __syncthreads();

    // Reduction to compute the sum within a block
    if (threadIdx.x == 0) {
        float blockSum = 0.0f;
        for (int i = 0; i < blockDim.x; ++i) {
            blockSum += temp[i];
        }
        atomicAdd(sum, blockSum); // Atomic add to avoid race conditions
    }
}