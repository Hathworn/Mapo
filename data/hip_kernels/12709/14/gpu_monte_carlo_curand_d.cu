#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_monte_carlo_curand_d(double *estimate, hiprandState *states) {
unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
int points_in_circle = 0;
double x, y;

hiprand_init(1234, tid, 0, &states[tid]);  // 	Initialize CURAND


for(int i = 0; i < TRIALS_PER_THREAD; i++) {
x = hiprand_uniform (&states[tid]);
y = hiprand_uniform (&states[tid]);
points_in_circle += (x*x + y*y <= 1.0f); // count if x & y is in the circle.
}
estimate[tid] = 4.0f * points_in_circle / (double) TRIALS_PER_THREAD; // return estimate of pi
}