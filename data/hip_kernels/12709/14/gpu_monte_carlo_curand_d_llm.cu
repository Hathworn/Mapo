#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gpu_monte_carlo_curand_d(double *estimate, hiprandState *states) {
    unsigned int tid = threadIdx.x + blockDim.x * blockIdx.x;
    int points_in_circle = 0;
    double x, y;

    // Use state update instead of reinitialization
    hiprandState localState = states[tid]; 

    // Unroll the loop for improved performance
    #pragma unroll
    for (int i = 0; i < TRIALS_PER_THREAD; i++) {
        x = hiprand_uniform(&localState);
        y = hiprand_uniform(&localState);
        points_in_circle += (x * x + y * y <= 1.0f);
    }

    // Store the updated state back
    states[tid] = localState; 

    estimate[tid] = 4.0f * points_in_circle / (double)TRIALS_PER_THREAD;
}