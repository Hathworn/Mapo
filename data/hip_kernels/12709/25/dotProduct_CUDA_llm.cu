#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dotProduct_CUDA(double *sum, long size, double *vector1, double *vector2) {
    long idx = blockIdx.x * blockDim.x + threadIdx.x; // Sequential thread index across threads
    if (idx < size) {
        // Perform element-wise multiplication directly
        sum[idx] = vector2[idx] * vector1[idx];
    }
}