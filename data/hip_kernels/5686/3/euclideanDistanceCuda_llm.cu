#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void euclideanDistanceCuda(float3* pDotProducts, size_t pSize, float* results) {
    int instance = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for cooperative fetching and reduced global memory access
    __shared__ float3 sharedDotProducts[1024]; // Adjust based on block size if needed

    while (instance < pSize) {
        sharedDotProducts[threadIdx.x] = pDotProducts[instance]; // Load into shared memory
        __syncthreads(); // Ensure all threads have loaded their data

        // Calculate result using fetched data
        float result = sharedDotProducts[threadIdx.x].x - 2 * sharedDotProducts[threadIdx.x].y + 
                        sharedDotProducts[threadIdx.x].z;
        
        // Single step for non-negativity constraint check
        results[instance] = fmaxf(result, 0.0f);

        // Increment by total number of threads
        instance += gridDim.x * blockDim.x;
        __syncthreads(); // Ensure all processing is complete before next iteration
    }
}