#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dotProductSingle(int* pFeatureList, float* pValuesList, size_t* pSizeOfInstanceList, size_t pSize, size_t pMaxNnz, float* pDevDotProduct) {
    int instanceId = blockIdx.x * blockDim.x + threadIdx.x;
    int threadId = threadIdx.x;
    extern __shared__ float value[];   // Use dynamic shared memory

    if (instanceId < pSize) {
        float temp = 0.0f;
        int jumpLength = instanceId * pMaxNnz;
        size_t size = pSizeOfInstanceList[instanceId];

        for (int i = threadId; i < size; i += blockDim.x) {
            float val = pValuesList[jumpLength + i];
            temp += val * val;
        }

        value[threadId] = temp;
        __syncthreads();

        // Reduction to accumulate results in shared memory
        for (int i = blockDim.x / 2; i > 0; i >>= 1) {
            if (threadId < i) {
                value[threadId] += value[threadId + i];
            }
            __syncthreads();
        }

        if (threadId == 0) {
            pDevDotProduct[instanceId] = value[0];
        }
    }
}