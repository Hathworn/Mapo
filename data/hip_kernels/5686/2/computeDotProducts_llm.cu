#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void computeDotProducts(float3* pDotProducts, size_t pSize, int* pCandidates, size_t* pJumpLength, size_t* pCandidateSize, 
                                   size_t pNumberOfCandidates, int* pFeatureIdsNeighbor, float* pValuesNeighbor, size_t pMaxNnzNeighbor, 
                                   size_t* pSizeNeighbor, int* pFeatureIdsInstance, float* pValuesInstance, size_t pMaxNnzInstance, 
                                   size_t* pSizeInstance, float* pPreComputedDotProductsNeighbor, float* pPreComputedDotProductsInstance) {

    int instanceCandidates = blockIdx.x;
    const int threadCount = 32;
    __shared__ int neighbor;
    __shared__ int instance;
    __shared__ int instanceCounter;
    __shared__ int pStartPosX;
    __shared__ int pEndPosX;
    __shared__ int pStartPosY;
    __shared__ int pEndPosY;

    float value = 0.0f; // Use register for value
    int featureIdX, featureIdY; // Use register space for these variables

    while (instanceCandidates < pNumberOfCandidates) {
        if (threadIdx.x == 0) {
            neighbor = pCandidates[pJumpLength[instanceCandidates]];
            instanceCounter = 0;
        }
        __syncthreads();
        while (instanceCounter < pCandidateSize[neighbor]) {
            if (threadIdx.x == 0) {
                instance = pCandidates[pJumpLength[instanceCandidates] + instanceCounter];
                pStartPosX = neighbor * pMaxNnzNeighbor;
                pEndPosX = neighbor * pMaxNnzNeighbor + pSizeNeighbor[neighbor];
                pStartPosY = instance * pMaxNnzInstance;
                pEndPosY = instance * pMaxNnzInstance + pSizeInstance[instance];
            }
            value = 0.0f; // Reset value for each instance
            __syncthreads();

            while (pStartPosX < pEndPosX && pStartPosY < pEndPosY) {
                featureIdX = pFeatureIdsNeighbor[pStartPosX + threadIdx.x];
                featureIdY = pFeatureIdsInstance[pStartPosY + threadIdx.x];

                if (featureIdX == featureIdY) {
                    value += pValuesNeighbor[pStartPosX + threadIdx.x] * pValuesInstance[pStartPosY + threadIdx.x];
                }
                __syncthreads();

                if (threadIdx.x == 0) {
                    // Handle boundary updates within the first thread
                    pStartPosX += threadCount;
                    pStartPosY += threadCount;
                }
                __syncthreads();
            }

            // Reduction to accumulate values
            for (int offset = warpSize / 2; offset > 0; offset /= 2) {
                value += __shfl_down_sync(0xffffffff, value, offset);
            }

            if (threadIdx.x == 0) {
                pDotProducts[pJumpLength[instanceCandidates] + instanceCounter].y = value;
                pDotProducts[pJumpLength[instanceCandidates] + instanceCounter].x = pPreComputedDotProductsNeighbor[neighbor];
                pDotProducts[pJumpLength[instanceCandidates] + instanceCounter].z = pPreComputedDotProductsInstance[instance];
                ++instanceCounter;
            }
            __syncthreads();
        }
        instanceCandidates += gridDim.x;
    }
}