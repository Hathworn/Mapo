#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "computeDotProducts.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    float3 *pDotProducts = NULL;
hipMalloc(&pDotProducts, XSIZE*YSIZE);
size_t pSize = XSIZE*YSIZE;
int *pCandidates = NULL;
hipMalloc(&pCandidates, XSIZE*YSIZE);
size_t *pJumpLength = NULL;
hipMalloc(&pJumpLength, XSIZE*YSIZE);
size_t *pCandidateSize = NULL;
hipMalloc(&pCandidateSize, XSIZE*YSIZE);
size_t pNumberOfCandidates = 1;
int *pFeatureIdsNeighbor = NULL;
hipMalloc(&pFeatureIdsNeighbor, XSIZE*YSIZE);
float *pValuesNeighbor = NULL;
hipMalloc(&pValuesNeighbor, XSIZE*YSIZE);
size_t pMaxNnzNeighbor = 1;
size_t *pSizeNeighbor = NULL;
hipMalloc(&pSizeNeighbor, XSIZE*YSIZE);
int *pFeatureIdsInstance = NULL;
hipMalloc(&pFeatureIdsInstance, XSIZE*YSIZE);
float *pValuesInstance = NULL;
hipMalloc(&pValuesInstance, XSIZE*YSIZE);
size_t pMaxNnzInstance = 1;
size_t *pSizeInstance = NULL;
hipMalloc(&pSizeInstance, XSIZE*YSIZE);
float *pPreComputedDotProductsNeighbor = NULL;
hipMalloc(&pPreComputedDotProductsNeighbor, XSIZE*YSIZE);
float *pPreComputedDotProductsInstance = NULL;
hipMalloc(&pPreComputedDotProductsInstance, XSIZE*YSIZE);
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
computeDotProducts<<<gridBlock, threadBlock>>>(pDotProducts,pSize,pCandidates,pJumpLength,pCandidateSize,pNumberOfCandidates,pFeatureIdsNeighbor,pValuesNeighbor,pMaxNnzNeighbor,pSizeNeighbor,pFeatureIdsInstance,pValuesInstance,pMaxNnzInstance,pSizeInstance,pPreComputedDotProductsNeighbor,pPreComputedDotProductsInstance);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
computeDotProducts<<<gridBlock, threadBlock>>>(pDotProducts,pSize,pCandidates,pJumpLength,pCandidateSize,pNumberOfCandidates,pFeatureIdsNeighbor,pValuesNeighbor,pMaxNnzNeighbor,pSizeNeighbor,pFeatureIdsInstance,pValuesInstance,pMaxNnzInstance,pSizeInstance,pPreComputedDotProductsNeighbor,pPreComputedDotProductsInstance);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
computeDotProducts<<<gridBlock, threadBlock>>>(pDotProducts,pSize,pCandidates,pJumpLength,pCandidateSize,pNumberOfCandidates,pFeatureIdsNeighbor,pValuesNeighbor,pMaxNnzNeighbor,pSizeNeighbor,pFeatureIdsInstance,pValuesInstance,pMaxNnzInstance,pSizeInstance,pPreComputedDotProductsNeighbor,pPreComputedDotProductsInstance);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}