#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize with shared memory and loop unrolling
__global__ void cosineSimilarityCuda(float3* pDotProducts, size_t pSize, float* results) {
    int instance = blockIdx.x * blockDim.x + threadIdx.x;

    __shared__ float3 sDotProducts[1024];
    sDotProducts[threadIdx.x] = pDotProducts[instance];
    __syncthreads();

    while (instance < pSize) {
        float3 dotProduct = sDotProducts[threadIdx.x];
        float sqrtX = sqrtf(dotProduct.x);
        float sqrtZ = sqrtf(dotProduct.z);
        results[instance] = dotProduct.y / (sqrtX * sqrtZ);
        instance += gridDim.x * blockDim.x;  // Ensure correct striding over grid dimensions
    }
}