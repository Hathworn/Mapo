#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_sparse_to_dense_assign(int n, const unsigned int *idx, float *src, float *trg) {
    // Use faster blockStride loop method for optimal memory access
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = id; i < n; i += stride) {
        trg[i] = src[idx[i]];
    }
}