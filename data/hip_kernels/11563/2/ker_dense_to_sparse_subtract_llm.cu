#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_dense_to_sparse_subtract(int n, const unsigned int *idx, float *src, float *trg) {
    // Utilize shared memory to optimize atomic operations
    __shared__ float cache[BLOCK_SIZE];
    int id = blockIdx.x*blockDim.x + threadIdx.x;

    if (id < n) {
        // Cache the subtraction result
        cache[threadIdx.x] = -src[id];
        __syncthreads();

        // Perform atomic addition with cached value
        if (threadIdx.x == 0) {
            for (int i = 0; i < blockDim.x; ++i) {
                atomicAdd(trg + idx[blockIdx.x * blockDim.x + i], cache[i]);
            }
        }
        __syncthreads();
    }
}