#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_sparse_to_dense_block_assign_and_multiply(int n, const unsigned *idx, int bsize, float mult, float* src, float *trg) {
    // Get the global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the block and element indices
    int block_id = id / bsize;
    int element_id = id % bsize;

    // Perform bounds check and assignment with multiplication
    if (block_id < n) {
        trg[id] = src[idx[block_id] * bsize + element_id] * mult;
    }
}