#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_dense_to_sparse_block_add(int n, const unsigned *idx, int bsize, float* src, float *trg) {
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure only valid threads process the data
    if (id < n * bsize) {
        // Pre-calculate indices for more efficient memory access
        int block_id = id / bsize;
        int offset = id % bsize;
        int target_index = idx[block_id] * bsize + offset;
        
        // Perform atomic addition to prevent race conditions
        atomicAdd(&trg[target_index], src[id]);
    }
}