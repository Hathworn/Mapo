#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void childKernel()
{
    // Use printf inside a block check to prevent clutter in block execution
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hello\n");
    }
}