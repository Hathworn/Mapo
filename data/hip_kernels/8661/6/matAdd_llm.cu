#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matAdd(int *yd, float *Ag, float *Bg, float *Cg) {
    // Calculate flattened index for 2D grid
    int idx = blockIdx.y * blockDim.y * (*yd) + blockIdx.x * blockDim.x + threadIdx.y * (*yd) + threadIdx.x;
    
    // Element-wise addition using flattened index
    Cg[idx] = Ag[idx] + Bg[idx];
}