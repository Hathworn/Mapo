#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void trace_subm(int j, int k, int *daG, int *dbG, double *AB, double *A) {
    int l;
    double sum = 0.0;  // Use a local variable to accumulate the sum
    int db = *dbG;    // Cache dereferenced pointers for better performance
    for (l = 0; l < db; l++) {
        sum += *(AB + j * db + l + k * db + l);
    }
    *(A + j * (*daG) + k) += sum;
}

__global__ void ptrBp(int *daG, int *dbG, double *ABg, double *Ag) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (k < *daG && j < *dbG) {  // Add boundary check to avoid out-of-bounds access
        trace_subm(j, k, daG, dbG, ABg, Ag);
    }
}