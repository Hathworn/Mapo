#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(int *xd, float *Ag, float *Bg, float *Cg) {
    // Calculate the global index for each thread
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Perform vector addition and store the result in Cg
    Cg[j] = Ag[j] + Bg[j] + *xd;
}