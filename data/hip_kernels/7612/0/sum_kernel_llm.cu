#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_kernel(float *g_odata, float *g_idata, int n)
{
    // size determined by the host application
    extern __shared__ float sdata[];

    // access thread id
    const unsigned int tid = threadIdx.x;
    const unsigned int idx = 2 * tid;

    // read input data and reduce collision by making accesses coalesced
    if (idx + 1 < n) {
        sdata[idx] = g_idata[idx];
        sdata[idx + 1] = g_idata[idx + 1];
    } else if (idx < n) {
        sdata[idx] = g_idata[idx];
        sdata[idx + 1] = 0.0f; // handle boundary case
    }

    __syncthreads();

    // perform computations more efficiently
    sdata[idx] += sdata[idx + 1];
    __syncthreads();

    if(tid < n/2) {
        g_odata[tid] = sdata[idx];
    }
}