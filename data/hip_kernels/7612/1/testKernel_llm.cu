#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void testKernel(float* g_idata, float* g_odata)
{
    // Use shared memory for intermediate results
    __shared__ float s_val1, s_val2;
    
    // Load global memory data into shared memory
    if (threadIdx.x == 0) {
        s_val1 = g_idata[0];
        s_val2 = g_idata[1];
    }
    __syncthreads();

    // Use registers for computation
    float result = 1.0f;
    float val1 = s_val1;
    float val2 = s_val2;

    // Perform some computations with unrolled loop
    #pragma unroll
    for (int i = 0; i < 4; ++i) { // Adjust loop count for actual workload
        result = val2 + (result * val1);
    }

    // Store result back to global memory from multiple threads reducing latency
    if (threadIdx.x == 0) {
        g_odata[0] = result;
    }
}