#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void NormalizeData(float* ShiftData, float MaxValue, float MinValue, int FinalDataSize)
{
    // Simplified thread index calculation for efficiency
    int id = blockIdx.x * blockDim.x + threadIdx.x 
           + blockIdx.y * gridDim.x * blockDim.x 
           + blockIdx.z * gridDim.x * gridDim.y * blockDim.x;

    // Optimized bounds checking
    if (id >= FinalDataSize) return;

    // Optimized normalization logic without branching
    float value = ShiftData[id];
    ShiftData[id] = value < MinValue ? 0 : (value > MaxValue ? 1 : (value - MinValue) / (MaxValue - MinValue));
}