#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void NormalizeData(float* ShiftData, float MaxValue, float MinValue, int FinalDataSize) {
    // Calculate global thread index more efficiently
    int id = blockIdx.x * blockDim.x * gridDim.y * gridDim.z + 
             blockIdx.y * blockDim.x * gridDim.z + 
             blockIdx.z * blockDim.x + 
             threadIdx.x;

    // Return early if out of bounds
    if (id >= FinalDataSize) return;

    // Normalize the data based on max/min value without printf for performance
    float value = ShiftData[id];
    if (value < MinValue) {
        ShiftData[id] = 0;
    } else if (value > MaxValue) {
        ShiftData[id] = 1;
    } else {
        ShiftData[id] = (value - MinValue) / (MaxValue - MinValue);
    }
}