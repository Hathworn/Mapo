#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void GetMatrixA(int* OCTData, float* MatrixA, int NumPolynomial, int OneDataSize)
{
    // Simple thread index computation
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early exit on ID exceeding boundaries
    if (id >= (NumPolynomial + 1) * (NumPolynomial + 1))
    {
        printf("Polynomial Fitting error!\n");
        return;
    }
    
    // Compute row and column indices
    int rowIndex = id % (NumPolynomial + 1);
    int colsIndex = id / (NumPolynomial + 1);
    
    // Reduce duplicate computation in loop
    float value = 0;
    float exponentRow = NumPolynomial - rowIndex;
    float exponentCol = NumPolynomial - colsIndex;
    
    for (int i = 0; i < OneDataSize; i++)
    {
        // Compute values once per iteration to avoid redundant operations
        float factor = (float)i / OneDataSize;
        value += pow(factor, exponentRow) * pow(factor, exponentCol);
    }
    
    // Store result in Matrix A
    MatrixA[id] = value;
}