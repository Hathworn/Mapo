#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void NormalizeData(float* ShiftData, float MaxValue, float MinValue, int FinalDataSize)
{
    // Calculate unique global thread ID in a simplified and performant way
    int id = blockIdx.x * blockDim.x + threadIdx.x + 
             blockIdx.y * blockDim.x * gridDim.x +
             blockIdx.z * blockDim.x * gridDim.x * gridDim.y;

    // Early exit if ID is out of bounds
    if (id >= FinalDataSize) return;

    // Simplifying normalization condition checks with ternary operator
    ShiftData[id] = (ShiftData[id] < MinValue) ? 0.0f : 
                    (ShiftData[id] > MaxValue) ? 1.0f :
                    (ShiftData[id] - MinValue) / (MaxValue - MinValue);
}