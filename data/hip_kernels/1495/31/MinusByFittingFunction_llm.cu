#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void MinusByFittingFunction(int* OCTData, float* PolyValue, int SizeZ)
{
    // Calculate unique global thread index using 3D block and grid dimensions.
    int id = (blockIdx.y * gridDim.x * gridDim.z + blockIdx.x * gridDim.z + blockIdx.z) * blockDim.x + threadIdx.x;

    // Extract the Z index using modulo operation for accessing PolyValue.
    int idZ = id % SizeZ;

    // Subtract predicted value from OCTData.
    OCTData[id] -= PolyValue[idZ];
}