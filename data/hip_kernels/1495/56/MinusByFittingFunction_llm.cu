#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void MinusByFittingFunction(int* OCTData, float* PolyValue, int SizeZ)
{
    // Calculate unique thread id for flattened 3D block and grid configuration
    int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x + 
             blockIdx.x * gridDim.z * blockDim.x + 
             blockIdx.z * blockDim.x + 
             threadIdx.x;

    // Ensure id is within bounds
    if (id < gridDim.x * gridDim.y * gridDim.z * blockDim.x) {
        // Extract the Z-index and adjust OCTData
        int idZ = id % SizeZ;
        OCTData[id] -= PolyValue[idZ];
    }
}