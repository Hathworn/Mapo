#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void ConnectPointsStatus(int* PointType_BestN, int* ConnectStatus, int size, int rows, int ChooseBestN, int ConnectRadius)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size * rows * ChooseBestN) 
        return;

    int sizeIndex = id / (rows * ChooseBestN);
    int tempID = id % (rows * ChooseBestN);
    int rowIndex = tempID / ChooseBestN;
    int chooseIndex = tempID % ChooseBestN;

    if (PointType_BestN[sizeIndex * rows * ChooseBestN + rowIndex * ChooseBestN + chooseIndex] == -1)
        return;
    
    int currentPoint = PointType_BestN[sizeIndex * rows * ChooseBestN + rowIndex * ChooseBestN + chooseIndex];
    int finalPos = min(rowIndex + ConnectRadius, rows);

    for (int i = rowIndex + 1; i < finalPos; i++) {
        int baseIndex = sizeIndex * rows * ChooseBestN + i * ChooseBestN;
        for (int j = 0; j < ChooseBestN; j++) {
            int nextPoint = PointType_BestN[baseIndex + j];
            if (nextPoint != -1) {
                int diffX = currentPoint - nextPoint;
                int diffY = i - rowIndex;
                int Radius = diffX * diffX + diffY * diffY;

                if (Radius < ConnectRadius * ConnectRadius) {
                    int index = sizeIndex * rows * ChooseBestN * ConnectRadius * ChooseBestN + 
                                rowIndex * ChooseBestN * ConnectRadius * ChooseBestN + 
                                chooseIndex * ConnectRadius * ChooseBestN + 
                                (i - rowIndex) * ChooseBestN + 
                                j;
                    ConnectStatus[index] = Radius;
                }
            }
        }
    }
}