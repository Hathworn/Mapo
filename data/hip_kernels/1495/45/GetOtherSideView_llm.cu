#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void GetOtherSideView(float* Data, float* OtherSideData, int SizeX, int SizeY, int FinalSizeZ)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early exit if the thread index is out of bounds
    if (id >= SizeX * SizeY) return;
    
    // Calculate row and column indices
    int idX = id / SizeY;
    int idY = id % SizeY;
    int DataOffsetIndex = idX * SizeY * FinalSizeZ + idY * FinalSizeZ;

    // Accumulate the sum of the Z dimension
    float totalZ = 0.0f;
    for (int i = 0; i < FinalSizeZ; i++)
    {
        totalZ += Data[DataOffsetIndex + i];
    }

    // Calculate the correct offset index and store the result
    int offsetIndex = idY * SizeX + idX;
    OtherSideData[offsetIndex] = totalZ;
}