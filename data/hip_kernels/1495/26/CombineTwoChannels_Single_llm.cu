#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void CombineTwoChannels_Single(int* OCTData_2Channls, int* OCTData, int SizeX, int SizeY, int SizeZ)
{
    // Calculate global thread id using optimized formula
    int id = blockIdx.x * blockDim.x + threadIdx.x + 
             blockIdx.y * gridDim.x * blockDim.x + 
             blockIdx.z * gridDim.x * gridDim.y * blockDim.x; 

    // Return if id is out of bounds
    if (id >= SizeX * SizeY * SizeZ) {
        printf("Combine Two Channel 有 Error!\n");
        return;
    }

    int BoxSize = SizeX * SizeZ;
    int BoxIndex = id / BoxSize;
    int BoxLeft = id % BoxSize;

    // Optimize data access by combining two channel values
    OCTData[id] = (OCTData_2Channls[BoxIndex * 2 * BoxSize + BoxLeft] +
                   OCTData_2Channls[(BoxIndex * 2 + 1) * BoxSize + BoxLeft]) / 2;
}