#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void CombineTwoChannels_Single(int* OCTData_2Channls, int* OCTData, int SizeX, int SizeY, int SizeZ)
{
    // Compute a unique global index for each thread
    int id = blockIdx.x * blockDim.x + threadIdx.x + 
             blockIdx.y * gridDim.x * blockDim.x + 
             blockIdx.z * gridDim.x * gridDim.y * blockDim.x;
    
    if (id >= SizeX * SizeY * SizeZ)
    {
        printf("Combine Two Channel error!\n");
        return;
    }
    
    int BoxSize = SizeX * SizeZ;
    int BoxIndex = id / BoxSize;
    int BoxLeft = id % BoxSize;
    
    // Optimize memory access patterns
    int idx1 = BoxIndex * 2 * BoxSize + BoxLeft;
    int idx2 = (BoxIndex * 2 + 1) * BoxSize + BoxLeft;
    
    OCTData[id] = (OCTData_2Channls[idx1] + OCTData_2Channls[idx2]) >> 1; // Use bitwise shift for division by 2
}