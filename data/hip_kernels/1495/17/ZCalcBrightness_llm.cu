#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void ZCalcBrightness(float* DataArray, float* BrightArray, int size, int rows, int cols, int startIndex) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size * rows)
        return;

    // Pre-calculate indices for optimized memory access
    int sizeIndex = id / rows;
    int rowIndex = id % rows;
    int baseIndex = sizeIndex * rows * cols + rowIndex * cols;

    // Initialize brightness value
    float brightness = 0.0f;

    // Apply loop unrolling for performance improvement
    int i;
    for (i = startIndex; i < cols - 4; i += 4) {
        brightness += DataArray[baseIndex + i] +
                      DataArray[baseIndex + i + 1] +
                      DataArray[baseIndex + i + 2] +
                      DataArray[baseIndex + i + 3];
    }
    for (; i < cols; i++) {
        brightness += DataArray[baseIndex + i];
    }

    // Store the result
    BrightArray[id] = brightness;
}