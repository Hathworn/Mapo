#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void GetMatrixA(int* OCTData, float* MatrixA, int NumPolynomial, int OneDataSize)
{
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if index exceeds expected bounds
    if (id >= (NumPolynomial + 1) * (NumPolynomial + 1))
    {
        printf("Polynomial Fitting issue!\n");
        return;
    }

    // Calculate row and column indices
    int rowIndex = id % (NumPolynomial + 1);
    int colsIndex = id / (NumPolynomial + 1);

    // Initialize accumulated value
    float value = 0.0f;

    // Precompute powers of index for performance
    int powerRow = NumPolynomial - rowIndex;
    int powerCol = NumPolynomial - colsIndex;

    // Sum across data points
    for (int i = 0; i < OneDataSize; i++)
    {
        // Directly use precomputed values to avoid repetitive work
        float normalized = (float)i / OneDataSize;
        float powerResult = pow(normalized, powerRow) * pow(normalized, powerCol);
        value += powerResult;
    }

    // Store result in MatrixA
    MatrixA[id] = value;
}