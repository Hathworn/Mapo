#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void CombineTwoChannels_Single(int* OCTData_2Channls, int* OCTData, int SizeX, int SizeY, int SizeZ)
{
    // Calculate global thread id directly using optimized formula
    int id = (blockIdx.z * gridDim.x * gridDim.y +
              blockIdx.y * gridDim.x +
              blockIdx.x) * blockDim.x + threadIdx.x;

    // Avoid out-of-bounds access
    if (id >= SizeX * SizeY * SizeZ) return;

    // Pre-computed BoxSize for performance improvement
    int BoxSize = SizeX * SizeZ;
    int BoxIndex = id / BoxSize;
    int BoxLeft = id % BoxSize;

    // Efficiently compute average and write to output
    int index1 = BoxIndex * 2 * BoxSize + BoxLeft;
    int index2 = (BoxIndex * 2 + 1) * BoxSize + BoxLeft;
    OCTData[id] = (OCTData_2Channls[index1] + OCTData_2Channls[index2]) / 2;
}