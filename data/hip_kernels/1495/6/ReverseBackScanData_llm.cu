#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void ReverseBackScanData(int* OCTData, int SizeX, int SizeY, int SizeZ)
{
    // Calculate unique global thread ID
    int id = (blockIdx.y * 2 + 1) * gridDim.x * 2 * gridDim.z * blockDim.x +
             blockIdx.x * gridDim.z * blockDim.x +
             blockIdx.z * blockDim.x +
             threadIdx.x;

    // Calculate mirrored thread ID for reverse operation
    int changeID = (blockIdx.y * 2 + 1) * gridDim.x * 2 * gridDim.z * blockDim.x +
                   (gridDim.y * 2 - blockIdx.x - 1) * gridDim.z * blockDim.x +
                   blockIdx.z * blockDim.x +
                   threadIdx.x;

    // Optimize swapping using a temporary variable
    int value = OCTData[id];
    OCTData[id] = OCTData[changeID];
    OCTData[changeID] = value;
}