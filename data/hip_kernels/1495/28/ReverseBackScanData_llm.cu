#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void ReverseBackScanData(int* OCTData, int SizeX, int SizeY, int SizeZ) {
    // Calculate thread index for forward and reverse elements with fewer operations
    int idx = threadIdx.x + blockDim.x * (blockIdx.z + gridDim.z * (blockIdx.x + gridDim.x * (blockIdx.y * 2 + 1)));
    int rdx = threadIdx.x + blockDim.x * (blockIdx.z + gridDim.z * ((gridDim.y * 2 - blockIdx.x - 1) + gridDim.x * (blockIdx.y * 2 + 1)));

    // Swap the values efficiently
    int temp = OCTData[idx];
    OCTData[idx] = OCTData[rdx];
    OCTData[rdx] = temp;
}