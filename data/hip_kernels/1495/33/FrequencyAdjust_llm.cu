#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void FrequencyAdjust(int* OCTData, float* KSpaceData, float* PXScale, int* IndexArray, int CutIndex, int SizeX, int SizeY, int SizeZ)
{
    int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +   // Calculate global thread index
             blockIdx.x * gridDim.z * blockDim.x +               //
             blockIdx.z * blockDim.x +                           //
             threadIdx.x;

    if (id >= SizeX * SizeY * SizeZ)
    {
        return; // Remove printf for efficiency
    }

    int idZ = id % SizeZ; // Calculate local Z index
    // Check conditions and initialize KSpaceData if needed
    if (IndexArray[idZ] == -1 || idZ >= CutIndex || idZ == 0)
    {
        KSpaceData[id] = 0;
        return;
    }

    // Compute previous PXScale index with clamp
    int LastPXScaleIndex = max(0, IndexArray[idZ] - 1); 

    // Simplified calculation for m and c
    double delta = PXScale[IndexArray[idZ]] - PXScale[LastPXScaleIndex];
    if (delta != 0) // Ensure division is valid
    {
        double m = (double)(OCTData[id] - OCTData[id - 1]) / delta;
        double c = OCTData[id] - m * PXScale[IndexArray[idZ]];
        KSpaceData[id] = m * idZ + c;
    }
    else
    {
        KSpaceData[id] = OCTData[id]; // Handle division by zero gracefully
    }
}