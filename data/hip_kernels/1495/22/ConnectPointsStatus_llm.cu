#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void ConnectPointsStatus(int* PointType_BestN, int* ConnectStatus, int size, int rows, int ChooseBestN, int ConnectRadius)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size * rows * ChooseBestN) // Check if id is out of bounds
        return;

    // Calculate indices
    int sizeIndex = id / (rows * ChooseBestN);
    int tempID = id % (rows * ChooseBestN);
    int rowIndex = tempID / ChooseBestN;
    int chooseIndex = tempID % ChooseBestN;

    // Check for invalid point
    if (PointType_BestN[sizeIndex * rows * ChooseBestN + rowIndex * ChooseBestN + chooseIndex] == -1)
        return;

    // Effective point processing
    int finalPos = min(rowIndex + ConnectRadius, rows);
    int currentPoint = PointType_BestN[sizeIndex * rows * ChooseBestN + rowIndex * ChooseBestN + chooseIndex];

    for (int i = rowIndex + 1; i < finalPos; i++)
    {
        for (int j = 0; j < ChooseBestN; j++)
        {
            // Get next point position
            int nextPoint = PointType_BestN[sizeIndex * rows * ChooseBestN + i * ChooseBestN + j];
            if (nextPoint != -1)
            {
                // Compute differences
                int diffX = currentPoint - nextPoint;
                int diffY = i - rowIndex;
                int Radius = diffX * diffX + diffY * diffY;

                // Check if within radius
                if (Radius < ConnectRadius * ConnectRadius)
                {
                    // Calculate index and update ConnectStatus
                    int index = sizeIndex * rows * ChooseBestN * ConnectRadius * ChooseBestN +
                                rowIndex * ChooseBestN * ConnectRadius * ChooseBestN +
                                chooseIndex * ConnectRadius * ChooseBestN +
                                (i - rowIndex) * ChooseBestN +
                                j;
                    
                    ConnectStatus[index] = Radius;
                }
            }
        }
    }
}