#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to compute MatrixA
__global__ static void GetMatrixA(int* OCTData, float* MatrixA, int NumPolynomial, int OneDataSize)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (id >= (NumPolynomial + 1) * (NumPolynomial + 1))
    {
        printf("Polynomial fitting issue!\n");
        return;
    }

    // Calculate indices
    int rowIndex = id % (NumPolynomial + 1);
    int colsIndex = id / (NumPolynomial + 1);

    // Accumulate values
    float value = 0.0f;
    float FirstValue, SecondValue;
    float polyRow = NumPolynomial - rowIndex;
    float polyCol = NumPolynomial - colsIndex;
    
    // Unroll loop for performance improvement
    for (int i = 0; i < OneDataSize; i += 4)
    {
        FirstValue = SecondValue = (float)i / OneDataSize;
        value += powf(FirstValue, polyRow) * powf(SecondValue, polyCol);
        
        if (i + 1 < OneDataSize)
        {
            FirstValue = SecondValue = (float)(i + 1) / OneDataSize;
            value += powf(FirstValue, polyRow) * powf(SecondValue, polyCol);
        }
        
        if (i + 2 < OneDataSize)
        {
            FirstValue = SecondValue = (float)(i + 2) / OneDataSize;
            value += powf(FirstValue, polyRow) * powf(SecondValue, polyCol);
        }
        
        if (i + 3 < OneDataSize)
        {
            FirstValue = SecondValue = (float)(i + 3) / OneDataSize;
            value += powf(FirstValue, polyRow) * powf(SecondValue, polyCol);
        }
    }
    
    MatrixA[id] = value;
}