#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function with unrolling and shared memory
__global__ static void ZCalcBrightness(float* DataArray, float* BrightArray, int size, int rows, int cols, int startIndex) 
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size * rows)
        return;

    __shared__ float sharedData[256]; // Utilize shared memory if possible

    int sizeIndex = id / rows;
    int rowIndex = id % rows;
    BrightArray[id] = 0;

    int stride = cols / 4; // Unroll loop to improve memory coalescing
    int remainder = cols % 4;
    
    for (int i = startIndex; i < startIndex + stride * 4; i += 4) 
    {
        int currentID = sizeIndex * rows * cols + rowIndex * cols + i;
        BrightArray[id] += DataArray[currentID]
                           + DataArray[currentID + 1]
                           + DataArray[currentID + 2]
                           + DataArray[currentID + 3];
    }
    
    // Handle remaining iterations if cols is not multiple of 4
    for (int i = startIndex + stride * 4; i < startIndex + stride * 4 + remainder; i++) 
    {
        int currentID = sizeIndex * rows * cols + rowIndex * cols + i;
        BrightArray[id] += DataArray[currentID];
    }
}