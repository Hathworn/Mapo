#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void ZCalcBrightness(float* DataArray, float* BrightArray, int size, int rows, int cols, int startIndex)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size * rows)  // Check bounds
        return;

    // Calculate Index
    int sizeIndex = id / rows;
    int rowIndex = id % rows;

    float sum = 0.0f;  // Use a local variable to reduce global memory access
    for (int i = startIndex; i < cols; i++)
    {
        int currentID = sizeIndex * rows * cols + rowIndex * cols + i;
        sum += DataArray[currentID];  // Accumulate results in the local variable
    }
    BrightArray[id] = sum;  // Store the final result to global memory
}