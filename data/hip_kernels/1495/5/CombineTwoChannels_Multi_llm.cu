#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ static void CombineTwoChannels_Multi(int* OCTData_2Channls, int* OCTData, int SizeX, int SizeY, int SizeZ)
{
    // Improved computation of global thread id for clarity
    int id = blockIdx.x + blockIdx.y * gridDim.x + blockIdx.z * gridDim.x * gridDim.y;
    id = id * blockDim.x + threadIdx.x;

    // Check bounds
    if (id >= SizeX * SizeY * SizeZ)
    {
        printf("Combine Two Channel 有 Error!\n");
        return;
    }

    // Compute offsets for channel combination
    int BoxSize = SizeX * SizeZ * 2;
    int BoxIndex = id / BoxSize;
    int BoxLeft = id % BoxSize;

    // Improved memory access pattern
    int index1 = BoxIndex * BoxSize * 2 + BoxLeft;
    int index2 = (BoxIndex * 2 + 1) * BoxSize + BoxLeft;
    
    // Combine channels
    OCTData[id] = (OCTData_2Channls[index1] + OCTData_2Channls[index2]) / 2;
}