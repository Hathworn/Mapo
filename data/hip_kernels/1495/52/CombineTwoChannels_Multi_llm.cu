#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void CombineTwoChannels_Multi(int* OCTData_2Channls, int* OCTData, int SizeX, int SizeY, int SizeZ)
{
    // Calculate the global thread ID more efficiently
    int id = blockIdx.x * blockDim.x + threadIdx.x +
             blockIdx.y * gridDim.x * blockDim.x +
             blockIdx.z * gridDim.x * gridDim.y * blockDim.x;

    if (id >= SizeX * SizeY * SizeZ) // Check if ID exceeds data range
    {
        printf("Combine Two Channel 有 Error!\n");
        return;
    }

    int BoxSize = SizeX * SizeZ * 2;
    int BoxIndex = id / BoxSize;
    int BoxLeft = id % BoxSize;

    // Perform computation with improved memory access logic
    int index1 = BoxIndex * 2 * BoxSize + BoxLeft;
    int index2 = (BoxIndex * 2 + 1) * BoxSize + BoxLeft;
    OCTData[id] = (OCTData_2Channls[index1] + OCTData_2Channls[index2]) / 2;
}