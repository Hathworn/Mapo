#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void ShiftFinalData(float* AfterFFTData, float* ShiftData, int SizeX, int SizeY, int FinalSizeZ, int FinalDataSize)
{
    // Calculate linear thread id
    int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x + 
             blockIdx.x * gridDim.z * blockDim.x + 
             blockIdx.z * blockDim.x + 
             threadIdx.x;

    // Ensure id is within bounds
    if (id >= FinalDataSize) return;

    // Calculate indices for Z, X, and Y
    int idZ = id % FinalSizeZ;
    int tempIndex = id / FinalSizeZ;
    int idX = tempIndex % SizeX;
    int idY = tempIndex / SizeX;

    // Wrap SizeY
    idY = (idY + SizeY / 2) % SizeY;

    // Compute new index
    int NewIndex = idY * SizeX * FinalSizeZ + idX * FinalSizeZ + idZ;

    // Assign shifted data
    ShiftData[id] = AfterFFTData[NewIndex];
}