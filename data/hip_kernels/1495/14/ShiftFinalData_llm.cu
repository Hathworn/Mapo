#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void ShiftFinalData(float* AfterFFTData, float* ShiftData, int SizeX, int SizeY, int FinalSizeZ, int FinalDataSize)
{
    // Calculate global thread ID
    int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +
             blockIdx.x * gridDim.z * blockDim.x +
             blockIdx.z * blockDim.x +
             threadIdx.x;

    // Early exit if out of bounds
    if (id >= FinalDataSize)
    {
        return;
    }

    // Calculate indices using modulo and division
    int idZ = id % FinalSizeZ;
    int tempIndex = id / FinalSizeZ;
    int idX = tempIndex % SizeX;
    int idY = tempIndex / SizeX;

    // Adjust idY by folding back
    idY = (idY + SizeY / 2) % SizeY;

    // Compute new index and assign shifted data
    int NewIndex = idY * SizeX * FinalSizeZ + idX * FinalSizeZ + idZ;
    ShiftData[id] = AfterFFTData[NewIndex];
}