#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void ShiftFinalData(float* AfterFFTData, float* ShiftData, int SizeX, int SizeY, int FinalSizeZ, int FinalDataSize)
{
    // Calculate global thread id
    int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x + 
             blockIdx.x * gridDim.z * blockDim.x + 
             blockIdx.z * blockDim.x + 
             threadIdx.x;

    // Early exit if id is out of bounds
    if (id >= FinalDataSize) return;

    // Compute indices directly to reduce computation
    int idZ = id % FinalSizeZ;
    int tempIndex = id / FinalSizeZ;
    int idX = tempIndex % SizeX;
    int idY = (tempIndex / SizeX + SizeY / 2) % SizeY;  // Adjust for SizeY folding

    // Calculate the new index and set shifted data
    int NewIndex = idY * SizeX * FinalSizeZ + idX * FinalSizeZ + idZ;
    ShiftData[id] = AfterFFTData[NewIndex];
}