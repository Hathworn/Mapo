#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void FrequencyAdjust(int* __restrict__ OCTData, float* __restrict__ KSpaceData, float* __restrict__ PXScale, int* __restrict__ IndexArray, int CutIndex, int SizeX, int SizeY, int SizeZ) {
    // Calculate global thread index with reduced arithmetic operations
    int id = threadIdx.x + blockDim.x * (blockIdx.z + gridDim.z * (blockIdx.x + gridDim.x * blockIdx.y));

    // Check if global thread index is within bounds
    if (id >= SizeX * SizeY * SizeZ) {
        printf("Frequency transformation issue encountered\n");
        return;
    }

    // Calculate idZ and check conditions early to avoid unnecessary computations
    int idZ = id % SizeZ;
    if (IndexArray[idZ] == -1 || idZ >= CutIndex || idZ == 0) {
        KSpaceData[id] = 0;
        return;
    }

    // Compute the slope (m) and intercept (c)
    int index = IndexArray[idZ];
    int previousIndex = max(index - 1, 0);
    double m = (double)(OCTData[id] - OCTData[id - 1]) / (PXScale[index] - PXScale[previousIndex]);
    double c = OCTData[id] - m * PXScale[index];

    // Calculate the adjusted frequency value and assign
    KSpaceData[id] = m * idZ + c;
}