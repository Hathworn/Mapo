#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void FrequencyAdjust(int* OCTData, float* KSpaceData, float* PXScale, int* IndexArray, int CutIndex, int SizeX, int SizeY, int SizeZ)
{
    // Calculate a global index using 3D grid and block dimensions
    int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +
             blockIdx.x * gridDim.z * blockDim.x +
             blockIdx.z * blockDim.x +
             threadIdx.x;

    // Boundary check to ensure id is within the data range
    if (id >= SizeX * SizeY * SizeZ)
    {
        printf("Frequency conversion issue detected");
        return;
    }

    // Calculate the Z index from the global id
    int idZ = id % SizeZ;

    // Conditions leading to zeroization of KSpaceData
    if (IndexArray[idZ] == -1 || idZ >= CutIndex || idZ == 0)
    {
        KSpaceData[id] = 0;
        return;
    }

    // Retrieve index of previous PXScale value, ensuring it is not negative
    int LastPXScaleIndex = max(IndexArray[idZ] - 1, 0);

    // Compute slope (m) and intercept (c) for linear equation
    double m = static_cast<double>(OCTData[id] - OCTData[id - 1]) / (PXScale[IndexArray[idZ]] - PXScale[LastPXScaleIndex]);
    double c = OCTData[id] - m * PXScale[IndexArray[idZ]];

    // Calculate adjusted KSpaceData using the linear equation
    KSpaceData[id] = m * idZ + c;
}