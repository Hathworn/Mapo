#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void ReverseBackScanData(int* OCTData, int SizeX, int SizeY, int SizeZ)
{
    // Calculate 1D index using Y, X, Z, and thread index
    int id = (blockIdx.y * 2 + 1) * gridDim.x * 2 * gridDim.z * blockDim.x +
             blockIdx.x * gridDim.z * blockDim.x +
             blockIdx.z * blockDim.x +
             threadIdx.x;

    // Calculate mirrored index for reverse operation.
    int changeID = (blockIdx.y * 2 + 1) * gridDim.x * 2 * gridDim.z * blockDim.x +
                   (gridDim.x * 2 - blockIdx.x - 1) * gridDim.z * blockDim.x +
                   blockIdx.z * blockDim.x +
                   threadIdx.x;

    // Swap values using atomic operations for potentially higher safety in concurrent environments.
    int value = atomicExch(&OCTData[id], OCTData[changeID]);
    atomicExch(&OCTData[changeID], value);
}