#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void GetOtherSideView(float* Data, float* OtherSideData, int SizeX, int SizeY, int FinalSizeZ)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Check within bounds
    if (id >= SizeX * SizeY) {
        printf("範圍有錯!!\n");
        return;
    }

    // Calculate indices
    int idX = id / SizeY;
    int idY = id % SizeY;
    int DataOffsetIndex = idX * SizeY * FinalSizeZ + idY * FinalSizeZ;

    // Calculate totalZ for the current id
    float totalZ = 0;
    for (int i = 0; i < FinalSizeZ; i++)
        totalZ += Data[DataOffsetIndex + i];
    
    // Compute and set OtherSideData offset
    int offsetIndex = idY * SizeX + idX;
    OtherSideData[offsetIndex] = totalZ;
}