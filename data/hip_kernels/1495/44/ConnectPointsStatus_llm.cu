#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void ConnectPointsStatus(int* PointType_BestN, int* ConnectStatus, int size, int rows, int ChooseBestN, int ConnectRadius)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size * rows * ChooseBestN)  
        return;

    // Precompute common expressions
    int sizeChooseBestN = size * rows * ChooseBestN;
    int sizeIndex = id / (rows * ChooseBestN);
    int tempID = id % (rows * ChooseBestN);
    int rowIndex = tempID / ChooseBestN;
    int chooseIndex = tempID % ChooseBestN;
    int baseIdx = sizeIndex * rows * ChooseBestN + rowIndex * ChooseBestN + chooseIndex;

    if (PointType_BestN[baseIdx] == -1)
        return;

    int connectRadiusSquared = ConnectRadius * ConnectRadius;
    int finalPos = min(rowIndex + ConnectRadius, rows);

    for (int i = rowIndex + 1; i < finalPos; i++) {
        for (int j = 0; j < ChooseBestN; j++) {

            int nextIdx = sizeIndex * rows * ChooseBestN + i * ChooseBestN + j;
            if (PointType_BestN[nextIdx] != -1) {
                // Calculate differences
                int diffX = PointType_BestN[baseIdx] - PointType_BestN[nextIdx];
                int diffY = i - rowIndex;
                int Radius = diffX * diffX + diffY * diffY;

                if (Radius < connectRadiusSquared) {
                    // Calculate index for ConnectStatus
                    int index = sizeIndex * rows * ChooseBestN * ConnectRadius * ChooseBestN +
                                rowIndex * ChooseBestN * ConnectRadius * ChooseBestN +
                                chooseIndex * ConnectRadius * ChooseBestN +
                                (i - rowIndex) * ChooseBestN +
                                j;
                    ConnectStatus[index] = Radius;
                }
            }
        }
    }
}