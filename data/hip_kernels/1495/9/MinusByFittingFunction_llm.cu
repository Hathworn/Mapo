#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void MinusByFittingFunction(int* OCTData, float* PolyValue, int SizeZ)
{
    // Calculate the unique global index for the thread
    int id = (blockIdx.y * gridDim.x * gridDim.z + blockIdx.x * gridDim.z + blockIdx.z) * blockDim.x + threadIdx.x;

    // Calculate the Z index
    int idZ = id % SizeZ;

    // Subtract the fitting value from the data
    OCTData[id] -= PolyValue[idZ];
}