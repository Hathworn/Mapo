#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void CombineTwoChannels_Multi(int* OCTData_2Channls, int* OCTData, int SizeX, int SizeY, int SizeZ)
{
    // Calculate global thread ID
    int id = blockIdx.y * gridDim.x * gridDim.z * blockDim.x +
             blockIdx.x * gridDim.z * blockDim.x +
             blockIdx.z * blockDim.x +
             threadIdx.x;

    // Check if the thread ID is out of bounds
    if (id >= SizeX * SizeY * SizeZ)
    {
        printf("Combine Two Channel 有 Error!\n");
        return;
    }

    int BoxSize = SizeX * SizeZ * 2;
    int BoxIndex = id / BoxSize;
    int BoxLeft = id % BoxSize;

    // Efficient memory access pattern
    int baseIndex1 = BoxIndex * 2 * BoxSize + BoxLeft;
    int baseIndex2 = (BoxIndex * 2 + 1) * BoxSize + BoxLeft;
    
    OCTData[id] = (OCTData_2Channls[baseIndex1] + OCTData_2Channls[baseIndex2]) / 2;
}