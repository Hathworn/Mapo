#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void GetOtherSideView(float* Data, float* OtherSideData, int SizeX, int SizeY, int FinalSizeZ)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= SizeX * SizeY)
    {
        return; // Remove unnecessary printf for efficiency
    }

    // Compute idX and idY
    int idX = id / SizeY;
    int idY = id % SizeY;
    int DataOffsetIndex = idX * SizeY * FinalSizeZ + idY * FinalSizeZ;

    // Sum up elements in the Z dimension
    float totalZ = 0.0f; // Initialize with 0.0f for consistency in floating-point arithmetic
    for (int i = 0; i < FinalSizeZ; i++)
    {
        totalZ += Data[DataOffsetIndex + i];
    }

    // Calculate and save result into OtherSideData
    int offsetIndex = idY * SizeX + idX;
    OtherSideData[offsetIndex] = totalZ;
}