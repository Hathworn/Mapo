#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void conv2genericrev(float *input, float *kernel, float *output, int input_n, int input_h, int input_w, int kernel_n, int kernel_h, int kernel_w, float alpha, int stride_h, int stride_w)
{
    // Output dimensions
    int output_h = input_h - (kernel_h - 1) * stride_h;
    int output_w = input_w - (kernel_w - 1) * stride_w;

    // Block and thread indices
    int kk = blockIdx.x;
    int ii = blockIdx.y;

    int batch = threadIdx.z;
    int kid = threadIdx.x;
    int nkids = blockDim.x;

    int tid = kid + batch * blockDim.x;
    int nthreads = blockDim.x * blockDim.z;

    // Pointer adjustment for output
    output = output + (kk * input_n + ii) * output_h * output_w; 

    // Shared memory for outputs
    extern __shared__ float shared_output[];

    // Shared memory index for this thread
    float *output_s = shared_output + tid * output_w * output_h;
    
    int yy = threadIdx.y;
    float *output_p = output_s + yy * output_w;

    for (int xx = 0; xx < output_w; xx++) {
        float *input_p = input + (ii + batch * input_n) * input_h * input_w + yy * stride_h * input_w + xx * stride_w;
        float *kernel_p = kernel + (kk + batch * kernel_n) * kernel_w * kernel_h;
        float sum = 0.0f;

        for (int ky = 0; ky < kernel_h; ky++) {
            for (int kx = kid; kx < kernel_w; kx += nkids) {
                sum += input_p[kx] * kernel_p[kx];
            }
            input_p += input_w;
            kernel_p += kernel_w;
        }
        output_p[xx] = sum;
    }
    
    __syncthreads();

    // Only one thread per 'yy' writes back results
    if (yy == 0) {
        // Perform reduction
        for (int k = 1; k < nthreads; k++) {
            for (int i = tid; i < output_w * output_h; i += nthreads) {
                shared_output[i] += shared_output[k * output_h * output_w + i];
            }
        }
        __syncthreads();

        // Add to existing output and write back
        for (int i = tid; i < output_w * output_h; i += nthreads) {
            output[i] += alpha * shared_output[i];
        }
    }
}