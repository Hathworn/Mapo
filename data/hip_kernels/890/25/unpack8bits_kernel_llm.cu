#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void unpack8bits_kernel(float *rcp, float *lcp, const int8_t *src) {
    // Calculate global index
    const size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t j = i * 2;
    
    // Optimize load by using a temporary variable
    int8_t src_j = src[j];
    int8_t src_j1 = src[j + 1];
    
    rcp[i] = static_cast<float>(src_j);
    lcp[i] = static_cast<float>(src_j1);
}