#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pfbFilter(float *filtered, float *unfiltered, float *taps, const int ntaps) {

    const int nfft = blockDim.x;
    const int i = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y;
    
    float sum = 0.0f; // Use a local variable for accumulation
    #pragma unroll
    for (int j = 0; j < ntaps; j++) {
        // Load data into registers to reduce global memory access
        float unfiltered_val = unfiltered[i + j * nfft];
        float tap_val = taps[threadIdx.x + j * nfft];
        sum += unfiltered_val * tap_val;
    }
    filtered[i] = sum; // Assign accumulated value to global memory
}