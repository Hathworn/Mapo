#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Accumulate(float4 *src, float4 *dest, int loop) {
    // Calculate global thread index
    const size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t k = blockDim.x * gridDim.x;

    // Initialize destination with the first source element
    float4 sum = src[i];

    // Unroll loop for better performance
    #pragma unroll
    for (int n = 1; n < loop; n++) {
        size_t index = i + n * k;
        sum.x += src[index].x;
        sum.y += src[index].y;
        sum.z += src[index].z;
        sum.w += src[index].w;
    }

    // Write the accumulated result
    dest[i] = sum;
}