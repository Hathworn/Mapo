#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void PowerInterleaved(float4 *src, float4 *dest) {
    // Calculate linear index
    const size_t i = blockDim.x * blockIdx.x + threadIdx.x;

    // Cache src values in registers to reduce global memory accesses
    float x = src[i].x;
    float y = src[i].y;
    float z = src[i].z;
    float w = src[i].w;

    // Cross pols
    dest[i].x += x * x + y * y;
    dest[i].y += z * z + w * w;
    // Parallel pols
    dest[i].z += x * z + y * w;
    dest[i].w += y * z - x * w;
}