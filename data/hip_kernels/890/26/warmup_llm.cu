#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *input, float *output) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform computation if within bounds (optional boundary check for safety)
    if (i < gridDim.x * blockDim.x) {
        output[i] = input[i] * input[i];
    }
}