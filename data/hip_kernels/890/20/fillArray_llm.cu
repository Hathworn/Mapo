#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fillArray(int8_t *dest, int loop) {
    const size_t i = blockDim.x * blockIdx.x + threadIdx.x;
    const size_t stride = blockDim.x * gridDim.x; // Use a more descriptive variable name for clarity

    float sineValue; // Precompute the sine function value to reduce redundant calculations
    for (int n = 0; n < loop; n++) {
        size_t index = i + n * stride; // Compute index only once per loop iteration
        sineValue = sin(index / 100.0f); // Use 100.0f for float precision
        dest[index] = sineValue * 30;
    }
}