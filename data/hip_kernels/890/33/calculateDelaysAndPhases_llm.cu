#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calculateDelaysAndPhases(double * gpuDelays, double lo, double sampletime, int fftsamples, int fftchannels, int samplegranularity, float * rotationPhaseInfo, int *sampleShifts, float* fractionalSampleDelays)
{
    size_t ifft = threadIdx.x + blockIdx.x * blockDim.x;
    size_t iant = blockIdx.y;
    int numffts = blockDim.x * gridDim.x;
    double meandelay, deltadelay, netdelaysamples_f, startphase;
    double d0, d2, a, b;
    double *interpolator = &(gpuDelays[iant * 4]);
    double filestartoffset = gpuDelays[iant * 4 + 3];
    float fractionaldelay;
    int netdelaysamples;

    // Calculate initial and ending delays for interpolation
    d0 = interpolator[0] * ifft * ifft + interpolator[1] * ifft + interpolator[2];
    d2 = interpolator[0] * (ifft + 1.0) * (ifft + 1.0) + interpolator[1] * (ifft + 1.0) + interpolator[2];

    // Calculate gradient and mean delay
    a = d2 - d0; // delay gradient
    b = d0; // start delay
    meandelay = a * 0.5 + b; // mean delay in the FFT window
    deltadelay = a / fftsamples; // delay change per sample

    netdelaysamples_f = (meandelay - filestartoffset) / sampletime;
    netdelaysamples = __double2int_rn(netdelaysamples_f / samplegranularity) * samplegranularity;

    // Store integer sample shifts
    sampleShifts[iant * numffts + ifft] = netdelaysamples;

    // Store fractional delay
    fractionaldelay = static_cast<float>(-(netdelaysamples_f - netdelaysamples) * 2 * M_PI / fftsamples);
    fractionalSampleDelays[iant * numffts + ifft] = fractionaldelay;

    // Calculate and store fringe rotation phases
    startphase = b * lo;
    rotationPhaseInfo[iant * numffts * 2 + ifft * 2] = static_cast<float>((startphase - int(startphase)) * 2 * M_PI);
    rotationPhaseInfo[iant * numffts * 2 + ifft * 2 + 1] = static_cast<float>(deltadelay * lo * 2 * M_PI);
}