#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pfbFilterShared(float *filtered, float *unfiltered, float *taps, const int ntaps) {
    extern __shared__ float shared_taps[];
    
    const int nfft = blockDim.x;
    const int i = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y;

    // Load taps into shared memory using coalesced access
    for (int j = threadIdx.x; j < ntaps * nfft; j += blockDim.x) {
        shared_taps[j] = taps[j];
    }
    __syncthreads();

    // Initialize filtered[i] with the first tap computation
    float temp = unfiltered[i] * shared_taps[threadIdx.x];
    for (int j = 1; j < ntaps; j++) {
        temp += unfiltered[i + j * nfft] * shared_taps[threadIdx.x + j * nfft];
    }
    filtered[i] = temp;
}