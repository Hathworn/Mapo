#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SegmentAllocLocInit(ushort2* gSegments, const uint32_t segmentCount)
{
    unsigned int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    if(globalId < segmentCount) // Directly check if within range
    {
        ushort2 defaultVal = {0xFFFF, 0xFFFF}; // Use a single default value assignment
        gSegments[globalId] = defaultVal; // More concise assignment
    }
}