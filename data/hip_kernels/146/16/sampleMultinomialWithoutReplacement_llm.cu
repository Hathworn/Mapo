#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;

        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithoutReplacement(hiprandStateMtgp32* state, int totalSamples, int sample, float* dest, long distributions, int categories, float* origDist, float* normDistPrefixSum) {
    // Optimize thread collaboration; reduce per-thread work
    const long threads_per_dist = blockDim.y;
    const long dist_per_block = gridDim.x * threads_per_dist;

    long curDistBase = blockIdx.x * threads_per_dist + threadIdx.y;
    while (curDistBase < distributions) {
        // Load random number only once per warp
        float r = hiprand_uniform(&state[blockIdx.x]);

        if (threadIdx.x == 0 && curDistBase < distributions) {
            int choice = binarySearchForMultinomial(normDistPrefixSum + curDistBase * categories, categories, r);
            dest[curDistBase * totalSamples + sample] = (float) choice + 1.0f;
            origDist[curDistBase * categories + choice] = 0.0f;
        }
        
        // Move to the next distribution for the block
        curDistBase += dist_per_block;
    }
}