#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int binarySearchForMultinomial(float* dist, int size, float val) {
    int start = 0;
    int end = size;

    while (end - start > 0) {
        int mid = start + (end - start) / 2;

        float midVal = dist[mid];
        if (midVal < val) {
            start = mid + 1;
        } else {
            end = mid;
        }
    }

    if (start == size) {
        // No probability mass or precision problems; just return the
        // first element
        start = 0;
    }

    return start;
}

__global__ void sampleMultinomialWithReplacement(hiprandStateMtgp32* state, int totalSamples, float* dest, long distributions, int categories, float* normDistPrefixSum) {
    // Use shared memory for reading prefix sums
    __shared__ float sharedPrefixSum[1024]; // Assumes categories <= 1024 for simplicity

    for (long curDist = blockIdx.x; curDist < distributions; curDist += gridDim.x) {
        // Load prefix sums into shared memory
        if (threadIdx.x < categories) {
            sharedPrefixSum[threadIdx.x] = normDistPrefixSum[curDist * categories + threadIdx.x];
        }
        __syncthreads(); // Ensure the prefix sums are loaded

        for (int sampleBase = blockIdx.y * blockDim.y; sampleBase < totalSamples; sampleBase += gridDim.y * blockDim.y) {
            int sample = sampleBase + threadIdx.y;

            // Use cooperative groups to manage synchronization within a warp
            if (sample < totalSamples) {
                float r = hiprand_uniform(&state[blockIdx.x * blockDim.y + threadIdx.y]);

                // Individual threads compute choices within the warp
                int choice = binarySearchForMultinomial(sharedPrefixSum, categories, r);

                if (threadIdx.x == 0) {
                    dest[curDist * totalSamples + sample] = (float)choice + 1.0f;
                }
            }
        }
        __syncthreads(); // Ensure all samples are processed before the next distribution
    }
}