#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexFill( float *tensor, long* stride, float *index, long src_nDim, int dim, long idx_size, long tensor_size, long size_dim, float val )
{
    int thread_idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;

    long flat_size = tensor_size / idx_size;

    if (thread_idx < flat_size)
    {
        // Cache values used in loops
        long index_stride_dim = stride[dim];

        for (int i = 0; i < idx_size; i++)
        {
            int leftover = thread_idx;
            int srcIdx = 0;
            // Combined loops for 'before' and 'after' dim
            for (int d = 0; d < src_nDim; d++)
            {
                // Avoid division when not necessary
                if (d == dim)
                {
                    continue;
                }
                coeff = leftover / stride[d];
                leftover -= coeff * stride[d];
                srcIdx += coeff * stride[d];
            }
            tensor[srcIdx + ((int)(index[i] - 1) * index_stride_dim)] = val;
        }
    }
}