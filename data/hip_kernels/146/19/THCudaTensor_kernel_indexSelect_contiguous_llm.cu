#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexSelect_contiguous(float *tensor, float *src, long stride, float *index, long idxSize)
{
    // Constants
    const int VT = 4;  // Vectorized load/store factor
    const int WARP_SIZE = 32;  // Number of threads in a warp

    // Calculate unique global thread index
    int idx = blockIdx.x * blockDim.y + threadIdx.y;
    int startIdx = threadIdx.x + blockIdx.y * VT * WARP_SIZE;

    // Optimize loop control and increments for larger strides
    for (; idx < idxSize; idx += blockDim.y * gridDim.x) {
        long srcIdx = static_cast<long>(index[idx]) - 1;
        srcIdx *= stride;
        long targetIdx = idx * stride;

        for (int offset = startIdx; offset < stride; offset += VT * WARP_SIZE * gridDim.y) {

            // Unroll load/store with better warp utilization
            #pragma unroll
            for (int i = 0; i < VT; i++) {
                int featureIdx = offset + i * WARP_SIZE;
                if (featureIdx < stride) {
                    tensor[targetIdx + featureIdx] = src[srcIdx + featureIdx];
                }
            }
        }
    }
}