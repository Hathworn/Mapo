#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_indexCopy(float *res, float *src, long *res_stride, float *index, long res_nDim, int dim, long idx_size, long src_size, long size_dim)
{
    // Use a more efficient calculation for thread index
    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the flat size outside the loop to reduce redundant operations
    long flat_size = src_size / idx_size;

    // Check if the thread index is within bounds
    if (thread_idx < flat_size)
    {
        // Precompute some common values outside of the loop
        int stride_dim = res_stride[dim];
        long target_incr = idx_size * stride_dim;

        for (int i = 0; i < idx_size; i++)
        {
            int leftover = thread_idx;
            int targetIdx = 0;
            int resIdx = 0;

            for (int d = 0; d < res_nDim; d++)
            {
                if (d < dim)
                {
                    long stride_d = res_stride[d] / size_dim;
                    int coeff = leftover / stride_d;
                    leftover -= coeff * stride_d;
                    targetIdx += coeff * stride_d * idx_size;
                    resIdx += coeff * res_stride[d];
                }
                else if (d > dim)
                {
                    int coeff = leftover / res_stride[d];
                    leftover -= coeff * res_stride[d];
                    targetIdx += coeff * res_stride[d];
                    resIdx += coeff * res_stride[d];
                }
            }
            int index_val = static_cast<int>(index[i]) - 1;
            res[resIdx + index_val * stride_dim] = src[targetIdx + i * stride_dim];
        }
    }
}