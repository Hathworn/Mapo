#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_copyUpperSymmetric(float *input, int n, int len)
{
    // Using grid-stride loop for improved access pattern and performance
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < len; idx += blockDim.x * gridDim.x) {
        const int r = idx % n;
        const int c = idx / n;
        if (r > c) {
            input[idx] = input[r * n + c];
        }
    }
}