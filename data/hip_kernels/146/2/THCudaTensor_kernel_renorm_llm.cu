#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void THCudaTensor_kernel_renorm(float *data, const float value, const long size, const float maxnorm)
{
    __shared__ float buffer[32];
    long tx = threadIdx.x;
    long bx = blockIdx.x;
    long step = blockDim.x;
    float *row = data + size * bx;

    buffer[tx] = 0;

    // Calculate the norm of the axis using parallel sum reduction
    for (long i = tx; i < size; i += step)
    {
        buffer[tx] += powf(fabsf(row[i]), value);
    }
    __syncthreads();

    // Using warp shuffle-based reduction for better efficiency
    for (int stride = warpSize / 2; stride > 0; stride /= 2)
    {
        buffer[tx] += __shfl_down_sync(0xFFFFFFFF, buffer[tx], stride);
    }
    if (tx % warpSize == 0) {
        atomicAdd(&buffer[0], buffer[tx]);
    }
    __syncthreads();

    // Clip norms and renormalize if necessary
    if (tx == 0) {
        float norm = powf(buffer[0], 1 / value);
        if (norm > maxnorm) {
            buffer[0] = maxnorm / (norm + 1e-7);
        } else {
            buffer[0] = 1.0f; // No renormalization needed
        }
    }
    __syncthreads();

    float renorm_factor = buffer[0];
    for (long i = tx; i < size; i += step)
    {
        row[i] *= renorm_factor;
    }
}