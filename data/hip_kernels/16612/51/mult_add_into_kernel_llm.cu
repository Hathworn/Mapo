#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Use 1D block and grid to simplify indexing and improve coalesced access
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds for memory access safety
    if(i < n){
        // Perform multiplication and addition directly
        c[i] += a[i] * b[i];
    }
}