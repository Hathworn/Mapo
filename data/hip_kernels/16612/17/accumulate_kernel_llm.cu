#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Use a register to accumulate sum
    float local_sum = 0.0f;
    for (int k = 0; k < n; ++k) {
        local_sum += x[k * groups + i];
    }
    sum[i] = local_sum;
}