#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified calculation of global index
    int stride = gridDim.x * blockDim.x; // Calculate the total number of threads

    for (; i < N; i += stride) { // Use stride loop for better workload distribution
        Y[i * INCY] *= X[i * INCX]; // Perform multiplication
    }
}