#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid stride loop for better utilization of GPU resources
    for (; i < n; i += blockDim.x * gridDim.x) {
        if (mask[i] == mask_num) x[i] *= scale;
    }
}