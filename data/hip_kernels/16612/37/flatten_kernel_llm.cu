#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a loop to handle multiple elements per thread
    while (i < N) {
        int in_s = i % spatial;
        int idx = i / spatial;
        int in_c = idx % layers;
        int b = idx / layers;

        int i1 = b * layers * spatial + in_c * spatial + in_s;
        int i2 = b * layers * spatial + in_s * layers + in_c;

        if (forward) out[i2] = x[i1];
        else out[i1] = x[i2];

        // Move to the next element by total number of threads
        i += blockDim.x * gridDim.x;
    }
}