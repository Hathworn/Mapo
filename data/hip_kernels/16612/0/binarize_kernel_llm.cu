#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Use blockIdx.z for computing global index, optimizing grid and block computation.
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = gridDim.x * blockDim.x;
    
    // Loop through all elements, handling boundary conditions.
    for (; i < n; i += gridSize) {
        binary[i] = (x[i] >= 0) ? 1 : -1;
    }
}