#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    
    // Exit if index is out of bounds
    if (i >= (NX + NY) * B) return;
    
    // Calculate batch and offset indices
    int b = i / (NX + NY);
    int j = i % (NX + NY);

    // Atomic addition for handling potential race conditions
    if (j < NX) {
        if (X) atomicAdd(&X[b * NX + j], OUT[i]);
    } else {
        if (Y) atomicAdd(&Y[b * NY + j - NX], OUT[i]);
    }
}