#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread index for the flattened grid.
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride to allow for larger number of threads handling more data.
    int stride = blockDim.x * gridDim.x;

    // Loop through all indices this thread is responsible for.
    for (; i < N; i += stride) {
        // Pre-calculate the index multiplication once per iteration.
        int idx = i * INCX;
        if ((X[idx] * X[idx]) < (ALPHA * ALPHA)) {
            X[idx] = 0;
        }
    }
}