#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Calculate unique index for thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    
    float sum = 0.0f; // Use local variable for reduction
    for(int j = 0; j < batch; ++j) {
        for(int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += delta[index] * (x[index] - mean[i]);
        }
    }

    // Avoid recomputation and use local variable
    float inv_var = -.5f * powf(variance[i] + 1e-5f, -1.5f);
    variance_delta[i] = sum * inv_var;
}