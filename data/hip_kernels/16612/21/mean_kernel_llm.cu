#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Thread index calculation
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    float scale = 1.f / (batch * spatial);

    // Ensure thread processes within bounds
    if (tid < filters) {
        float sum = 0.0f;

        // Unroll inner loops for coalesced memory access
        for (int j = 0; j < batch; ++j) {
            for (int k = 0; k < spatial; ++k) {
                int index = j * filters * spatial + tid * spatial + k;
                sum += x[index];
            }
        }
        
        mean[tid] = sum * scale;  // Store scaled mean result
    }
}