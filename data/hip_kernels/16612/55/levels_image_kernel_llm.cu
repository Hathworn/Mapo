#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x;
    float s = hsv.y;
    float v = hsv.z;

    float r, g, b;
    float f, p, q, t;

    if (s == 0) {
        r = g = b = v;
    } else {
        int index = (int) floorf(h);
        f = h - index;
        p = v * (1.f - s);
        q = v * (1.f - s * f);
        t = v * (1.f - s * (1.f - f));
        switch (index) {
            case 0: r = v, g = t, b = p; break;
            case 1: r = q, g = v, b = p; break;
            case 2: r = p, g = v, b = t; break;
            case 3: r = p, g = q, b = v; break;
            case 4: r = t, g = p, b = v; break;
            default: r = v, g = p, b = q; break;
        }
    }
    r = fminf(fmaxf(r, 0.f), 1.f);
    g = fminf(fmaxf(g, 0.f), 1.f);
    b = fminf(fmaxf(b, 0.f), 1.f);
    return make_float3(r, g, b);
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x;
    float g = rgb.y;
    float b = rgb.z;

    float h, s, v;
    float max = fmaxf(r, fmaxf(g, b));
    float min = fminf(r, fminf(g, b));
    float delta = max - min;
    v = max;
    if (max == 0) {
        s = 0;
        h = -1;
    } else {
        s = delta / max;
        if (r == max) {
            h = (g - b) / delta;
        } else if (g == max) {
            h = 2 + (b - r) / delta;
        } else {
            h = 4 + (r - g) / delta;
        }
        h = (h < 0) ? h + 6 : h;
    }
    return make_float3(h, s, v);
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;

    // Calculate positions
    int x = id % w;
    int y = (id / w) % h;
    id /= (w * h);

    // Pre-fetch random values
    float rshift = rand[0];
    float gshift = rand[1];
    float bshift = rand[2];
    float r0 = rand[8 * id + 0];
    float r1 = rand[8 * id + 1];
    float r2 = rand[8 * id + 2];
    float r3 = rand[8 * id + 3];

    // Adjust saturation and exposure
    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > 0.5f) ? 1.f / saturation : saturation;
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > 0.5f) ? 1.f / exposure : exposure;

    // Offset image pointer
    size_t offset = id * h * w * 3;
    image += offset;

    // Load RGB values from image
    float r = image[x + w * (y + h * 0)];
    float g = image[x + w * (y + h * 1)];
    float b = image[x + w * (y + h * 2)];
    float3 rgb = make_float3(r, g, b);

    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }

    // Write modified RGB values back to image
    image[x + w * (y + h * 0)] = rgb.x * scale + translate + (rshift - 0.5f) * shift;
    image[x + w * (y + h * 1)] = rgb.y * scale + translate + (gshift - 0.5f) * shift;
    image[x + w * (y + h * 2)] = rgb.z * scale + translate + (bshift - 0.5f) * shift;
}