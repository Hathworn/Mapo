#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N) { // Optimize guard check by moving inside the main logic
        int f = (index / spatial) % filters;
        // Precompute reused values for efficiency
        float var_f_inv_sqrt = rsqrtf(variance[f] + .00001f);
        float spatial_batch = spatial * batch;
        float x_mean_diff = x[index] - mean[f];

        // Use of precomputed values
        delta[index] = delta[index] * var_f_inv_sqrt 
                     + variance_delta[f] * 2.f * x_mean_diff / spatial_batch 
                     + mean_delta[f] / spatial_batch;
    }
}