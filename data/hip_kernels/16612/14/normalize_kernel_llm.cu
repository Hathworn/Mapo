#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread index is within bounds
    if (index < N) {
        int f = (index / spatial) % filters;
        
        // Perform normalization
        x[index] = (x[index] - mean[f]) / (sqrtf(variance[f] + .00001f));
    }
}