#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate thread's global index efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop through data using grid-stride loop to cover entire array
    for(; i < N; i += gridDim.x * blockDim.x) {
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}