#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float* x, float* delta, float* mean, float* variance, int batch, int filters, int spatial, float* variance_delta) 
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Loop unrolling to reduce the number of iterations
    for (int j = 0; j < batch; ++j) {
        for (int i = 0; i < spatial; i += threads * 2) {
            int index1 = j * spatial * filters + filter * spatial + i + id;
            int index2 = index1 + threads;

            // Use conditional assignment to avoid conditional statements
            float delta_contribution1 = (i + id < spatial) ? delta[index1] * (x[index1] - mean[filter]) : 0;
            float delta_contribution2 = (i + id + threads < spatial) ? delta[index2] * (x[index2] - mean[filter]) : 0;

            local[id] += delta_contribution1 + delta_contribution2;
        }
    }

    __syncthreads();

    // Use a single thread to reduce the array
    if (id == 0) {
        variance_delta[filter] = 0;
        for (int i = 0; i < threads; ++i) {
            variance_delta[filter] += local[i];
        }
        variance_delta[filter] *= -.5f * powf(variance[filter] + .00001f, -1.5f);
    }
}