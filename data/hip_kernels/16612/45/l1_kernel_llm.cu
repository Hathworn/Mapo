#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Optimize by skipping re-calculation of global index if using 1D grid (in most common cases)
    if(i < n){
        float diff = truth[i] - pred[i];
        error[i] = fabsf(diff); // use intrinsic fabsf for performance
        delta[i] = copysignf(1.0f, diff); // use copysignf function for cleaner logic
    }
}