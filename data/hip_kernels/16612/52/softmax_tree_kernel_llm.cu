#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    float sum = 0;
    float largest = -INFINITY;
    
    // Find the largest element in the input array
    for (int i = 0; i < n; ++i) {
        float val = input[i * stride];
        largest = fmaxf(val, largest);
    }
    
    // Calculate exponentials and sum them
    for (int i = 0; i < n; ++i) {
        float e = expf((input[i * stride] - largest) / temp);
        sum += e;
        output[i * stride] = e;
    }
    
    // Normalize the output array
    for (int i = 0; i < n; ++i) {
        output[i * stride] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int total = spatial * batch * groups;
    
    // Process each element only if within bounds
    while (id < total) {
        int s = id % spatial;
        int tmp_id = id / spatial;
        int g = tmp_id % groups;
        int b = tmp_id / groups;
        int goff = group_offset[g] * spatial;
        int boff = b * stride;
        softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);

        id += blockDim.x * gridDim.x; // Move by grid size for next element
    }
}