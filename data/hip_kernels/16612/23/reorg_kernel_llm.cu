#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Use a single index calculation to improve coherence
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return; // Ensure no out-of-bounds access

    // Precompute division results to avoid redundant calculations
    int b = index / (w * h * c);
    int residual = index % (w * h * c);
    int in_c = (residual / (w * h)) % c;
    int in_h = (residual / w) % h;
    int in_w = residual % w;

    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;

    int out_index = b * (out_c * w * h * stride * stride) + c2 * (w * stride * h * stride) + h2 * (w * stride) + w2;

    // Use ternary operator for concise code
    out[forward ? out_index : index] = x[forward ? index : out_index];
}