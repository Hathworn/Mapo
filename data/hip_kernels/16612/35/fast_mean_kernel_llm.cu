#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0;

    // Reduce global memory access frequency by unrolling loops
    int step = spatial / threads * threads;  // handle remaining elements separately
    for(int j = 0; j < batch; ++j) {
        for(int i = 0; i < step; i += threads){
            int index = j * spatial * filters + filter * spatial + i + id;
            local[id] += x[index];
        }
        // Handle remaining elements if spatial is not divisible by threads
        for(int i = step + id; i < spatial; i += threads){
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += x[index];
        }
    }

    __syncthreads();

    // Use efficient reduction pattern
    if (id < threads / 2){
        local[id] += local[id + threads / 2];
    }
    __syncthreads();
    
    if(id < threads / 4){
        local[id] += local[id + threads / 4];
    }
    __syncthreads();
    
    if(id < threads / 8){
        local[id] += local[id + threads / 8];
    }
    __syncthreads();
    
    if(id == 0){
        mean[filter] = local[0] / (spatial * batch);
    }
}