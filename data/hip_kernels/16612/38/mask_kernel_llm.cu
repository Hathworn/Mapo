#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    // Calculate unique thread index in grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread does not access out-of-bounds memory and apply mask
    if (i < n && mask[i] == mask_num)
    {
        x[i] = val;
    }
}