#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Compute global index optimally
    int gridStride = gridDim.x * blockDim.x; // Add grid stride for loop optimization
    for (; i < (NX+NY)*B; i += gridStride) { // Use stride loop for better performance
        int b = i / (NX+NY);
        int j = i % (NX+NY);
        OUT[i] = (j < NX) ? X[b*NX + j] : Y[b*NY + j - NX]; // Use ternary operator for condition check
    }
}