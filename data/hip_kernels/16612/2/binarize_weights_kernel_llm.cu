#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    // Calculate global thread index
    int f = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Ensure the thread operates on a valid element
    if (f >= n) return;

    // Use shared memory to reduce global memory accesses for mean calculation
    extern __shared__ float shared_weights[];
    float mean = 0.0f;

    // Each thread sums up its portion of the weights into shared memory
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[threadIdx.x] = fabsf(weights[f * size + i]);
        __syncthreads();

        // Perform reduction to calculate mean
        for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
            if (threadIdx.x < offset) {
                shared_weights[threadIdx.x] += shared_weights[threadIdx.x + offset];
            }
            __syncthreads();
        }

        // Calculate mean from reduced sum at the first thread
        if (threadIdx.x == 0) {
            mean = shared_weights[0] / size;
        }
        __syncthreads();
    }
    
    // Apply thresholding to binarize the weights
    for(int i = threadIdx.x; i < size; i += blockDim.x) {
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }
}