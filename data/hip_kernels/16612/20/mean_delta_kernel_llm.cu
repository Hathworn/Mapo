#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread within filters
    if (i >= filters) return;

    // Initialize mean_delta for filter i
    float temp_mean_delta = 0.0f;

    // Compute mean_delta for filter i
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            temp_mean_delta += delta[index];
        }
    }

    // Update global memory for mean_delta[i]
    mean_delta[i] = temp_mean_delta * (-1.f / sqrtf(variance[i] + .00001f));
}