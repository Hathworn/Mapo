#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[BLOCK];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0.0f;

    // Unroll loop to improve performance
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += powf((x[index] - mean[filter]), 2);
        }
    }

    __syncthreads();

    // Reduce the shared memory array to compute variance
    if(id < 32) {
        for (int i = threads / 2; i > 0; i >>= 1) {
            if (id < i) {
                local[id] += local[id + i];
            }
            __syncthreads();
        }
    }

    if (id == 0) {
        variance[filter] = local[0] / (spatial * batch - 1);
    }
}