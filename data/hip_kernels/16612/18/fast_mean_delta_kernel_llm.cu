#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0.0f;

    // Precompute shared memory contribution
    for (int j = 0; j < batch; ++j) {
        for (int i = id; i < spatial; i += threads) {
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += delta[index];
        }
    }

    __syncthreads();

    // Use parallel reduction to sum up local[id] within a block
    for (int stride = threads / 2; stride > 0; stride /= 2) {
        if (id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    // Write final result to mean_delta
    if (id == 0) {
        mean_delta[filter] = -local[0] / sqrtf(variance[filter] + 0.00001f);
    }
}