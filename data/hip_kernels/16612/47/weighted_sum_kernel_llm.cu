#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate global thread index in 1D array
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Process only if within bounds
    if(i < n) {
        // Use ternary operator to avoid branching
        float b_value = b ? b[i] : 0;
        c[i] = s[i] * a[i] + (1 - s[i]) * b_value;
    }
}