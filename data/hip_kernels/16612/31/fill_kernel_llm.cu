#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use block dimension stride to iterate through array
    for (; i < N; i += blockDim.x * gridDim.x)
    {
        X[i * INCX] = ALPHA; // Set value
    }
}