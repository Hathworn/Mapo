#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use of strided loops for better memory access pattern
    int stride = blockDim.x * gridDim.x;
    for (int idx = index; idx < n * size * batch; idx += stride) {
        int i = idx % size;
        int j = (idx / size) % n;
        int k = idx / (size * n);

        output[(k * n + j) * size + i] += biases[j];
    }
}