#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop through elements in the grid stride loop
    for(; i < N; i += gridDim.x * blockDim.x)
    {
        // Constrain each element
        X[i * INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i * INCX]));
    }
}