#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use shared memory to load a tile of data 
    __shared__ float tile[256]; 

    // Compute a unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop over segments of data
    for (int index = i; index < N; index += blockDim.x * gridDim.x)
    {
        // Load data into shared memory
        tile[threadIdx.x] = X[index * INCX];

        // Synchronize to ensure all threads have written to shared memory
        __syncthreads();

        // Perform the addition on data in shared memory
        tile[threadIdx.x] += ALPHA;
        
        // Ensure all threads have completed computation
        __syncthreads();

        // Store the result back to global memory
        X[index * INCX] = tile[threadIdx.x];
    }
}