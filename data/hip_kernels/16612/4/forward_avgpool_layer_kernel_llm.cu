#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int k = id % c;
    int b = id / c;
    int out_index = k + c * b;
    int in_start_index = h * w * (k + b * c);
    int in_end_index = in_start_index + w * h;
    
    // Optimize: Accumulate using thread-local variable
    float sum = 0.0f;
    for (int i = in_start_index; i < in_end_index; ++i) {
        sum += input[i];
    }
    
    // More efficient to divide just once
    output[out_index] = sum / (w * h);
}