#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplify index calculation
    if (index >= n) return;
    
    float sum = 0;
    for(int b = 0; b < batch; ++b) {
        int i = b * n + index;
        sum += delta[i];
    }
    atomicAdd(&bias_updates[index], sum); // Use atomicAdd for thread safety
}