#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out) 
{
    // Calculate global thread index
    size_t i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (i >= N) return;

    // Precompute factors for index conversion
    int wc_stride = w * stride;
    int hc_stride = h * stride;
    int wh = w * h;

    // Convert flat index to multidimensional indices
    int out_w = i % wc_stride;
    int out_h = (i / wc_stride) % hc_stride;
    int out_c = (i / (wc_stride * hc_stride)) % c;
    int b = (i / (wc_stride * hc_stride * c)) % batch;

    // Compute input indices
    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_index = b * wh * c + out_c * wh + in_h * w + in_w;

    // Forward or backward operation
    if(forward) {
        // Forward pass: update output
        out[i] += scale * x[in_index];
    } else {
        // Backward pass: atomic add
        atomicAdd(x + in_index, scale * out[i]);
    }
}