#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Optimize with stride in inner loop for better memory coalescing
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Reduce sum using shared memory, parallelized reduction
    if (p == 0) {
        float block_sum = 0;
        for (int i = 0; i < BLOCK; ++i) {
            block_sum += part[i];
        }
        atomicAdd(&bias_updates[filter], block_sum); // Use atomic add for thread-safe update
    }
}