#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    // Calculate the global index for the thread
    int s = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (s >= size) return;

    // Initialize mean variable and use shared memory to reduce global memory accesses
    extern __shared__ float shared_mean[];
    shared_mean[threadIdx.x] = 0.0f;

    // Compute the mean of absolute values in parallel
    for (int i = 0; i < n; ++i){
        shared_mean[threadIdx.x] += fabsf(input[i * size + s]);
    }

    // Reduce the sum in shared memory to calculate the mean
    float mean = 0.0f;
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        __syncthreads();
        if (threadIdx.x < offset) {
            shared_mean[threadIdx.x] += shared_mean[threadIdx.x + offset];
        }
    }
    if (threadIdx.x == 0) {
        mean = shared_mean[0] / n;
        shared_mean[0] = mean;
    }

    // Broadcast the computed mean to all threads in the block
    __syncthreads();
    mean = shared_mean[0];

    // Binarize the input based on the computed mean
    for (int i = 0; i < n; ++i){
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}