#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function to apply scale and bias
__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int offset = blockIdx.x * blockDim.x + threadIdx.x;
    int filter = blockIdx.y;
    int batch = blockIdx.z;
    
    if(offset < size) {
        // Perform calculations using registers for efficiency
        int index = (batch * n + filter) * size + offset;
        float bias = biases[filter];
        output[index] *= bias;
    }
}