#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define THREADS_PER_BLOCK 256  // Define optimal threads per block

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
  extern __shared__ float shared_mem[];  // Use shared memory for more efficient reduction
  float* shared_largest = shared_mem;
  float* shared_exp_sum = shared_mem + blockDim.x;
  
  int i = threadIdx.x;
  float sum = 0;
  float largest = -INFINITY;

  // Find the largest value for numerical stability
  if (i < n) {
    int idx = i * stride;
    float val = input[idx];
    atomicMax(&shared_largest[0], val);  // Use atomicMax for reduction
  }
  __syncthreads();
  largest = shared_largest[0];

  // Compute exponentials
  if (i < n) {
    int idx = i * stride;
    float e = expf((input[idx] - largest)/temp);
    output[idx] = e;
    atomicAdd(&shared_exp_sum[0], e);  // Use atomicAdd for sum reduction
  }
  __syncthreads();
  sum = shared_exp_sum[0];

  // Normalize outputs
  if (i < n) {
    int idx = i * stride;
    output[idx] /= sum;
  }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  if (id >= batch * groups) return;
  int b = id / groups;
  int g = id % groups;
  
  softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
}