#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    // Pre-calculate constants outside loop
    float one_minus_powB1 = 1.f - powf(B1, t);
    float one_minus_powB2 = 1.f - powf(B2, t);

    // Use pre-calculated constants to optimize division operations
    float mhat = m[index] / one_minus_powB1;
    float vhat = v[index] / one_minus_powB2;

    // More efficient calculation of x
    x[index] += rate * mhat / (sqrtf(vhat) + eps);
}