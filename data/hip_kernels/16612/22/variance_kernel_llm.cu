#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    // Precompute constants
    float scale = 1.f / (batch * spatial - 1);

    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (i >= filters) return;
    
    float varSum = 0.0f;  // Use a local variable to accumulate the variance for better performance

    // Use a single loop with flattened index access
    for (int j = 0; j < batch * spatial; ++j) {
        int index = j * filters + i;
        float diff = x[index] - mean[i];
        varSum += diff * diff;
    }

    // Write the computed variance to the output
    variance[i] = varSum * scale;
}