#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f;

    // Prefetch indices to optimize global memory access
    int size_filter_n = size * n;
    int offset = size_filter_n * batch;
    for(int b = 0; b < batch; ++b) {
        for(int i = p; i < size; i += BLOCK) {
            // Load data if within bounds
            int index = i + size * (filter + n * b);
            if (index < offset) {
                sum += delta[index] * x_norm[index];
            }
        }
    }
    part[p] = sum;
    __syncthreads();

    // Reduction within the block
    if (p == 0) {
        float block_sum = 0.0f;
        for(int i = 0; i < BLOCK; ++i) {
            block_sum += part[i];
        }
        atomicAdd(&scale_updates[filter], block_sum); // Use atomic add to avoid collision
    }
}