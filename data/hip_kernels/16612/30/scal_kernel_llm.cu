#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate thread's unique global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop through all elements using stride
    while (i < N) {
        X[i * INCX] *= ALPHA;
        i += blockDim.x * gridDim.x;  // Move to the next element within thread's range
    }
}