#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Calculate the global index for the current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    // Utilize only necessary threads for computation
    if(i < N) {
        // Precompute index values to minimize repetitive calculations
        int X_index = i * INCX;
        int Y_index = i * INCY;
        // Optimize power operation 
        Y[Y_index] = powf(X[X_index], ALPHA);
    }
}