#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    if (i < n) {
        float t = truth[i];
        float p = pred[i];
        float log_p = (p > 0) ? __logf(p) : 0;  // Fast logarithm approximation
        error[i] = t ? -log_p : 0;
        delta[i] = t - p;
    }
}