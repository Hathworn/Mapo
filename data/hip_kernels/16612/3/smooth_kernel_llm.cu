#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    // Compute global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    // Calculate indices
    int j = id % w;
    int i = (id / w) % h;
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    // Calculate output index
    int out_index = j + w * (i + h * (k + c * b));

    // Calculate offset
    int offset = size / 2;

    for(int l = -offset; l <= offset; ++l){
        int cur_h = i + l;
        for(int m = -offset; m <= offset; ++m){
            int cur_w = j + m;
            if(cur_h >= 0 && cur_h < h && cur_w >= 0 && cur_w < w){
                int index = cur_w + w * (cur_h + h * (k + b * c));
                // Update delta with valid elements
                delta[out_index] += rate * (x[index] - x[out_index]);
            }
        }
    }
}