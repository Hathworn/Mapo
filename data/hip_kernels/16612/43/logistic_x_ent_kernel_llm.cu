#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation
    if (i < n) {
        float t = truth[i];
        float p = pred[i];
        float log_p = log(p + 1e-7f);               // Small epsilon for numerical stability
        float log_1_p = log(1.0f - p + 1e-7f);     // Small epsilon for numerical stability
        error[i] = -t * log_p - (1 - t) * log_1_p;
        delta[i] = t - p;
    }
}