#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel for better memory access patterns and reduced branching
__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(i < n){
        float dc_i = dc[i];
        float s_i = s[i];
        ds[i] += dc_i * (a[i] - b[i]);

        // Use conditional within the same block to reduce branching
        if (da) {
            float da_i = dc_i * s_i;
            da[i] += da_i;
        }
        if (db) {
            float db_i = dc_i * (1.0f - s_i);
            db[i] += db_i;
        }
    }
}