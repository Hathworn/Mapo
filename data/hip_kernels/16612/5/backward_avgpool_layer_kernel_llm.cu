#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n * c) return; // Combined bounds check

    int k = id % c;
    int b = id / c;

    int out_index = k + c * b;
    int base_in_index = h * w * (k + b * c); // Precompute the base input index

    // Unroll the loop for better performance
    for (int i = 0; i < w * h; i += 1) {
        int in_index = base_in_index + i;
        in_delta[in_index] += out_delta[out_index] / (w * h);
    }
}