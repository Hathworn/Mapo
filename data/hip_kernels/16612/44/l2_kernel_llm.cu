#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if(i < n){
        float diff = truth[i] - pred[i];
        error[i] = diff * diff; // I know this is technically wrong, deal with it.
        delta[i] = diff;
    }
}