#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int area = (size - 1) / stride;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return; // Terminate if id exceeds n

    int index = id;
    int j = index % in_w;
    int i = (index / in_w) % in_h;
    int k = (index / (in_w * in_h)) % in_c;
    int b = index / (in_w * in_h * in_c);

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    float d = 0;
    // Combine area loops to reduce condition checks
    for (int l = 0; l <= size - stride; l += stride) {
        for (int m = 0; m <= size - stride; m += stride) {
            int out_w = (j - w_offset + m) / stride;
            int out_h = (i - h_offset + l) / stride;
            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h) {
                int out_index = out_w + w * (out_h + h * (k + in_c * b));
                if (indexes[out_index] == index) {
                    d += delta[out_index];
                }
            }
        }
    }
    prev_delta[index] += d; // Update prev_delta
}