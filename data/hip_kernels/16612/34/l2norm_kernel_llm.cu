#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    int index = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (index >= N) return;
    int b = index / spatial;
    int i = index % spatial;
    
    float sum = 0.0f;

    // Unroll loop for parallel reduction and efficiency
    for (int f = 0; f < filters; ++f) {
        int idx = b*filters*spatial + f*spatial + i;
        sum += x[idx] * x[idx];
    }

    sum = sqrtf(sum);
    if (sum == 0.0f) sum = 1.0f;

    for (int f = 0; f < filters; ++f) {
        int idx = b*filters*spatial + f*spatial + i;
        x[idx] /= sum;
        dx[idx] = (1.0f - x[idx]) / sum;
    }
}