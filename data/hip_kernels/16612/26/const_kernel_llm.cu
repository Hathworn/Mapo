#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use a more concise expression to calculate the index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop for better parallel workload distribution
    for (; i < N; i += blockDim.x * gridDim.x) 
    {
        X[i * INCX] = ALPHA;
    }
}