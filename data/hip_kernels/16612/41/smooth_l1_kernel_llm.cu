#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error) {
    // Optimized index calculation for better readability
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure all threads are within bounds
    if (i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);

        // Optimize conditional statements using ternary operator
        error[i] = (abs_val < 1) ? (diff * diff) : (2 * abs_val - 1);
        delta[i] = (abs_val < 1) ? diff : ((diff > 0) ? 1 : -1);
    }
}