#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate global index using optimized method
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process multiple elements per thread
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        Y[OFFY + idx * INCY] += ALPHA * X[OFFX + idx * INCX];
    }
}