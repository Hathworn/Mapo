#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta) {
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    // Utilize shared memory for partial results to reduce global memory accesses
    __shared__ float shared_sum[32];
    __shared__ float shared_norm1[32];
    __shared__ float shared_norm2[32];

    float sum = 0.0f;
    float norm1 = 0.0f;
    float norm2 = 0.0f;

    for (int b = 0; b < batch; ++b) {
        for (int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            float val1 = output[i1];
            float val2 = output[i2];
            sum += val1 * val2;
            norm1 += val1 * val1;
            norm2 += val2 * val2;
        }
    }

    // Use atomic operations to accumulate results in shared memory
    atomicAdd(&shared_sum[threadIdx.x % 32], sum);
    atomicAdd(&shared_norm1[threadIdx.x % 32], norm1);
    atomicAdd(&shared_norm2[threadIdx.x % 32], norm2);

    // Synchronize threads to ensure shared memory is correctly updated
    __syncthreads();

    // Use thread 0 of each block to finalize the computation
    if (threadIdx.x % 32 == 0) {
        float norm1_final = sqrt(shared_norm1[threadIdx.x]);
        float norm2_final = sqrt(shared_norm2[threadIdx.x]);
        float norm = norm1_final * norm2_final;
        float sum_final = shared_sum[threadIdx.x] / norm;

        for (int b = 0; b < batch; ++b) {
            for (int i = 0; i < size; ++i) {
                int i1 = b * size * n + f1 * size + i;
                int i2 = b * size * n + f2 * size + i;
                delta[i1] += -scale * sum_final * output[i2] / norm;
                delta[i2] += -scale * sum_final * output[i1] / norm;
            }
        }
    }
}