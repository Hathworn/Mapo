#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// __syncthreads() is used to synchronize threads within a block
__global__ void sync_conv_groups() {
    // Synchronize all threads in the block
    __syncthreads();
}