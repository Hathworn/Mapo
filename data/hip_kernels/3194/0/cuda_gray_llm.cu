#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_gray(unsigned char *input, int offset, int streamSize, unsigned char* gray, int size) {

    // Calculate thread index once and use it for both rgb_idx and gray_idx
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int gray_idx = (offset / 3) + tid;
    int rgb_idx = offset + (tid * 3);

    if (rgb_idx >= streamSize || gray_idx >= size) {
        return;
    }

    // Optimize memory access by storing input values in registers
    unsigned char r = input[rgb_idx];
    unsigned char g = input[rgb_idx + 1];
    unsigned char b = input[rgb_idx + 2];

    gray[gray_idx] = (gray_value[0] * r) + (gray_value[1] * g) + (gray_value[2] * b);
}