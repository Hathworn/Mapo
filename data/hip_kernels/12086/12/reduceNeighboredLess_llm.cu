#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighboredLess(int *g_idata, int *g_odata, unsigned int n) {
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if (idx >= n) return;

    // Intra-block reduction using shared memory
    __shared__ int sdata[1024];  // Assuming blockDim.x <= 1024
    sdata[tid] = idata[tid];
    __syncthreads();

    // In-place reduction
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}
```
