#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, int NX, int NY)
{
    // Calculate the global thread index
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure threads do not access out-of-bounds memory
    if (ix < NX && iy < NY)
    {
        unsigned int idx = iy * NX + ix;
        // Perform addition operation
        C[idx] = A[idx] + B[idx];
    }
}