#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling4(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 4;

    // Unrolling 4
    int temp_sum = 0;
    if (idx + 3 * blockDim.x < n)
    {
        temp_sum = g_idata[idx] + g_idata[idx + blockDim.x]
                 + g_idata[idx + 2 * blockDim.x] + g_idata[idx + 3 * blockDim.x];
    }
    idata[tid] = temp_sum;

    // Synchronize threads before reduction
    __syncthreads();

    // Shared memory reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads(); // Synchronize within threadblock
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}