#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceInterleaved (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    extern __shared__ int sdata[];
    sdata[tid] = (idx < n) ? g_idata[idx] : 0;
    __syncthreads();

    // perform reduction using shared memory to minimize global memory accesses
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            sdata[tid] += sdata[tid + stride];
        }
        
        __syncthreads();
    }

    // write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}