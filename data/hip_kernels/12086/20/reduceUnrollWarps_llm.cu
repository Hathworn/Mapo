#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrollWarps(int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // Unrolling 2
    if (idx + blockDim.x < n) {
        g_idata[idx] += g_idata[idx + blockDim.x];
    }

    __syncthreads();

    // In-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1)
    {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // Unrolling last warp efficiently
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        #pragma unroll // Unroll for better performance
        for (int offset = 32; offset > 0; offset >>= 1) {
            vsmem[tid] += vsmem[tid + offset];
        }
    }

    // Thread 0 writes the result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = idata[0];
    }
}