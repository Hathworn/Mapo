#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;  // Use blockDim.x for flexible thread block size
    const int y = blockIdx.y * blockDim.y + threadIdx.y;  // Use blockDim.y for flexible thread block size
    if (x < width && y < height) {  // Check bounds before computation
        int sz = height * pitch;
        int p = threadIdx.z * sz + y * pitch + x;
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
}