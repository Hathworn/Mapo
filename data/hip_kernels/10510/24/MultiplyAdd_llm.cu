#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    // Calculate the global thread index
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;

    // Ensure valid index range
    if (x < width && y < height)
    {
        // Perform multiply-add operation
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
    // Remove __syncthreads() as it is unnecessary here
}