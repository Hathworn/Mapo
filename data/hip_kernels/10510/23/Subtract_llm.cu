#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Calculate x and y using blockDim instead of hardcoding 16
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;

    // Check bounds within valid range
    if (x < width && y < height)
    {
        // Perform subtraction only if within bounds
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
    // Remove unnecessary __syncthreads as no shared memory is used
}