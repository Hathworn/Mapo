#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int pitch, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x*128 + tx;
    const int y = blockIdx.y*16;
    int p = y*pitch + x;

    // Load initial values into shared memory
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
    } else {
        minvals[tx] = FLT_MAX; // Ensure no influence from invalid index
        maxvals[tx] = -FLT_MAX;
    }
    
    // Reduce within the block to find local min and max
    int stride = pitch;
    for (int ty = 1; ty < 16; ty++, p += stride) {
        if (x < width) {
            float val = d_Data[p];
            minvals[tx] = fminf(minvals[tx], val);
            maxvals[tx] = fmaxf(maxvals[tx], val);
        }
    }
    __syncthreads();

    // Parallel reduction to find global block min and max
    for (int offset = blockDim.x/2; offset > 0; offset >>= 1) {
        if (tx < offset) {
            minvals[tx] = fminf(minvals[tx], minvals[tx + offset]);
            maxvals[tx] = fmaxf(maxvals[tx], maxvals[tx + offset]);
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
}