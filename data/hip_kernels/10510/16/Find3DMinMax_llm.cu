#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Find3DMinMax(int *d_Result, float *d_Data1, float *d_Data2, float *d_Data3, int width, int pitch, int height) {
    // Data cache
    __shared__ float data1[3 * (MINMAX_W + 2)];
    __shared__ float data2[3 * (MINMAX_W + 2)];
    __shared__ float data3[3 * (MINMAX_W + 2)];
    __shared__ float ymin1[(MINMAX_W + 2)];
    __shared__ float ymin2[(MINMAX_W + 2)];
    __shared__ float ymin3[(MINMAX_W + 2)];
    __shared__ float ymax1[(MINMAX_W + 2)];
    __shared__ float ymax2[(MINMAX_W + 2)];
    __shared__ float ymax3[(MINMAX_W + 2)];

    // Current tile and apron limits, relative to row start
    const int tx = threadIdx.x;
    const int xStart = blockIdx.x * MINMAX_W;
    const int xEnd = xStart + MINMAX_W - 1;
    const int xReadPos = xStart + tx - WARP_SIZE;
    const int xWritePos = xStart + tx;
    const int xEndClamped = min(xEnd, width - 1);
    int memWid = MINMAX_W + 2;

    int memPos0 = (tx - WARP_SIZE + 1);
    int memPos1 = (tx - WARP_SIZE + 1);
    int yq = 0;
    unsigned int output = 0;

    for (int y = 0; y < 32 + 2; y++) {
        output >>= 1;
        int memPos = yq * memWid + (tx - WARP_SIZE + 1);
        int yp = 32 * blockIdx.y + y - 1;
        yp = max(yp, 0);
        yp = min(yp, height - 1);
        int readStart = yp * pitch;

        // Load data into shared memory if within bounds
        if (tx >= (WARP_SIZE - 1)) {
            if (xReadPos < 0 || xReadPos >= width) {
                data1[memPos] = 0;
                data2[memPos] = 0;
                data3[memPos] = 0;
            } else {
                data1[memPos] = d_Data1[readStart + xReadPos];
                data2[memPos] = d_Data2[readStart + xReadPos];
                data3[memPos] = d_Data3[readStart + xReadPos];
            }
        }
        __syncthreads();

        int memPos2 = yq * memWid + tx;
        if (y > 1) {
            if (tx < memWid) {
                // Minimize redundant function calls by using temporary variables
                float val1_0 = data1[memPos0], val1_1 = data1[memPos1], val1_2 = data1[memPos2];
                float val2_0 = data2[memPos0], val2_1 = data2[memPos1], val2_2 = data2[memPos2];
                float val3_0 = data3[memPos0], val3_1 = data3[memPos1], val3_2 = data3[memPos2];

                float min1 = fminf(fminf(val1_0, val1_1), val1_2);
                float min2 = fminf(fminf(val2_0, val2_1), val2_2);
                float min3 = fminf(fminf(val3_0, val3_1), val3_2);
                float max1 = fmaxf(fmaxf(val1_0, val1_1), val1_2);
                float max2 = fmaxf(fmaxf(val2_0, val2_1), val2_2);
                float max3 = fmaxf(fmaxf(val3_0, val3_1), val3_2);

                ymin1[tx] = min1;
                ymin2[tx] = fminf(fminf(min1, min2), min3);
                ymin3[tx] = min3;
                ymax1[tx] = max1;
                ymax2[tx] = fmaxf(fmaxf(max1, max2), max3);
                ymax3[tx] = max3;
            }
        }
        __syncthreads();

        if (y > 1) {
            if (tx < MINMAX_W) {
                if (xWritePos <= xEndClamped) {
                    // Reduce the number of fminf and fmaxf calls
                    float minVal = fminf(fminf(ymin2[tx], ymin2[tx + 2]), fminf(fminf(ymin1[tx + 1], ymin3[tx + 1]), fminf(data2[memPos0 + 1], data2[memPos2 + 1])));
                    float maxVal = fmaxf(fmaxf(ymax2[tx], ymax2[tx + 2]), fmaxf(fmaxf(ymax1[tx + 1], ymax3[tx + 1]), fmaxf(data2[memPos0 + 1], data2[memPos2 + 1])));
                    
                    minVal = fminf(minVal, d_Threshold[1]);
                    maxVal = fmaxf(maxVal, d_Threshold[0]);

                    // Check data2[memPos1 + 1] against thresholds
                    if (data2[memPos1 + 1] < minVal || data2[memPos1 + 1] > maxVal) {
                        output |= 0x80000000;
                    }
                }
            }
        }
        __syncthreads();

        memPos0 = memPos1;
        memPos1 = memPos2;
        yq = (yq < 2 ? yq + 1 : 0);
    }

    // Write output if within width bounds
    if (tx < MINMAX_W && xWritePos < width) {
        int writeStart = blockIdx.y * pitch + xWritePos;
        d_Result[writeStart] = output;
    }
}