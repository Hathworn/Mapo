#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    // Calculate global thread coordinates
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Linear index based on 2D grid coordinates
    int p = y * pitch + x;

    // Check boundaries
    if (x < width && y < height) {
        // Perform element-wise subtraction
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}