#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;

    // Coalesce memory accesses and improving iteration efficiency
    for (int y = 0; y < h; y += 32)
    {
        for (int x = 0; x < w; x += 16)
        {
            unsigned int val = minmax[((y / 32) * w) + x + tx];
            int base = y * w + x + tx;
            if (val)
            {
                // Loop unrolling to reduce overhead
                for (int k = 0; k < 32; k += 4)
                {
                    #pragma unroll
                    for (int i = 0; i < 4; ++i)
                    {
                        if ((val & 0x1) && (numPts < maxPts))
                        {
                            ptrs[16 * numPts + tx] = base + (k + i) * w;
                            numPts++;
                        }
                        val >>= 1;
                    }
                }
            }
        }
    }
}