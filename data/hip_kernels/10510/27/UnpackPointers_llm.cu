#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    const int blockYOffset = blockIdx.y * 32; // Compute block offset for y
    int numPts = 0;

    if (blockYOffset < h) {
        for (int x = 0; x < w; x += 16) {
            unsigned int val = minmax[blockYOffset * w + x + tx];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        // Compute global index directly
                        ptrs[16 * numPts + tx] = (blockYOffset + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
        }
    }
}