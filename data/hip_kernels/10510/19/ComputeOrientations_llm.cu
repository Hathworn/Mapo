#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h)
{
    __shared__ float data[16*15];
    __shared__ float hist[32*13];
    __shared__ float gauss[16];
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;

    // Initialize histogram bins to zero
    for (int i = 0; i < 13; i++) {
        hist[i*32 + tx] = 0.0f;
    }
    __syncthreads();

    // Pre-compute Gaussian values
    float i2sigma2 = -1.0f / (2.0f * 3.0f * 3.0f);
    if (tx < 15) {
        gauss[tx] = exp(i2sigma2 * (tx - 7) * (tx - 7));
    }

    // Fetch location from pointers
    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;

    // Pre-calculate px and corresponding x value
    int px = xp & 15;
    int x = tx - px;

    // Load data into shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = max(0, min(w - 1, xp + x));
        int yi = max(0, min(h - 1, yp + y));

        if (x >= 0 && x < 15) {
            data[memPos] = g_Data[yi * w + xi];
        }
    }
    __syncthreads();

    // Compute gradients and vote into orientation histogram
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = __float2int_rn(16.0f * atan2f(dy, dx) / 3.1416f + 16.5f);  // Avoids if-condition, uses rounding
            bin = (bin == 32) ? 0 : bin;  // Correct bin wrapping
            float grad = sqrtf(dx * dx + dy * dy);
            atomicAdd(&hist[32 * (x - 1) + bin], grad * gauss[x] * gauss[y]);  // Atomic add to prevent race condition
        }
    }
    __syncthreads();

    // Reduction to base histogram
    for (int stride = 8; stride > 0; stride /= 2) {
        if (tx < stride) {
            hist[tx] += hist[tx + stride];
        }
        __syncthreads();
    }

    // Smoothing histogram using a sliding kernel
    if (tx < 32) {
        float v1 = hist[(tx + 1) & 31];
        float v2 = hist[(tx + 31) & 31];
        hist[tx] = 6 * hist[tx] + 4 * (v1 + v2) + (hist[(tx + 2) & 31] + hist[(tx + 30) & 31]);
    }
    __syncthreads();

    // Find peaks in the histogram
    if (tx == 0) {
        float maxval1 = 0.0f;
        float maxval2 = 0.0f;
        int i1 = -1;
        int i2 = -1;
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }

        // Compute first orientation peak
        float val1 = hist[32 + ((i1 + 1) & 31)];
        float val2 = hist[32 + ((i1 + 31) & 31)];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Orient[bx] = 11.25f * ((peak < 0.0f) ? peak + 32.0f : peak);

        // Compute second orientation peak
        if (maxval2 < 0.8f * maxval1) {
            i2 = -1;
        }
        if (i2 >= 0) {
            val1 = hist[32 + ((i2 + 1) & 31)];
            val2 = hist[32 + ((i2 + 31) & 31)];
            peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Orient[bx + maxPts] = 11.25f * ((peak < 0.0f) ? peak + 32.0f : peak);
        } else {
            d_Orient[bx + maxPts] = i2;
        }
    }
}