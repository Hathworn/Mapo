#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_int(int* x, int leng) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Check index bounds before accessing array
    if (i < leng) {
        printf("%d,", x[i]);
    }
}