#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_double(double* x, int leng) {
    // Using grid stride loop for better parallel execution
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < leng; i += blockDim.x * gridDim.x) {
        printf("%lf,", x[i]);
    }
}