#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_float(float* x, int leng) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure valid index and remove redundant array access
    if (i < leng) {
        float value = x[i];  // Cache value to reduce global memory access
        printf("%f,", value); // Print cached value
    }
}