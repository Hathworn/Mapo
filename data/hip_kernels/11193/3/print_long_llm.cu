#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void print_long(long* x, int leng) {
    // Cached blockDim.x for improved performance
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds to prevent out of range errors
    if (idx < leng) {
        printf("%ld,", x[idx]);
    }
}