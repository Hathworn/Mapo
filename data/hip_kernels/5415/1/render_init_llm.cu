#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Return early for out-of-bound threads
    if (i >= max_x || j >= max_y) return;
    
    // Calculate the linear pixel index
    int pixel_index = j * max_x + i;
    
    // Initialize random state with unique seed for each pixel
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}