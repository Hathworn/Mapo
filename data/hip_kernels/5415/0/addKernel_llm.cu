#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

// Optimized kernel with grid-stride loop
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global index
    int stride = blockDim.x * gridDim.x; // Calculate stride for grid-stride loop
    for (; i < size; i += stride) {
        c[i] = a[i] + b[i];
    }
}