#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_sparse_to_dense_block_assign_and_multiply(int n, const unsigned *idx, int bsize, float mult, float* src, float *trg) {
    // Optimized global thread ID calculation
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure in-bounds access and operate efficiently
    if (id < n * bsize) {
        int base_idx = idx[id / bsize] * bsize; // Precompute for efficiency
        trg[id] = src[base_idx + id % bsize] * mult;
    }
}