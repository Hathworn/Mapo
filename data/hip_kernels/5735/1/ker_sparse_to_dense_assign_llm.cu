#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_sparse_to_dense_assign(int n, const unsigned int *idx, float *src, float *trg) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < n) {
        // Use registers to store intermediate values which might improve performance slightly
        unsigned int srcIdx = idx[id];
        float value = src[srcIdx];
        trg[id] = value;
    }
}