#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_dense_to_sparse_block_add(int n, const unsigned *idx, int bsize, float* src, float *trg) {
    // Get global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we are within bounds for processing
    if (id < n * bsize) {
        // Calculate target index and perform atomic addition for concurrency safety
        atomicAdd(&trg[idx[id / bsize] * bsize + id % bsize], src[id]);
    }
}