#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_dense_to_sparse_subtract(int n, const unsigned int *idx, float *src, float *trg) {
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds before atomic operation
    if (id < n) {
        // Subtract src[id] from trg[idx[id]] using atomic operation
        atomicAdd(&trg[idx[id]], -src[id]);
    }
}