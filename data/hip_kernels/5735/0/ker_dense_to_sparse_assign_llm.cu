#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_dense_to_sparse_assign(int n, const unsigned int *idx, float *src, float *trg) {
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Early return optimization to avoid branching
    if (id >= n) return;

    // Direct assignment for in-bounds thread
    trg[idx[id]] = src[id];
}