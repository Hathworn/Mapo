#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define IMUL(a, b) __mul24(a, b)
#define iDivUp(a,b) (((a)+(b)-1)/(b))
#define CONV1_THREAD_SIZE 256
#define CONVN_THREAD_SIZE1 16
#define CONVN_THREAD_SIZE2 31

#define nrFilters 28
#define nrDirs 8

__constant__ float d_v1popDirs[3][nrFilters] = {
    // Directions data...
};

__constant__ float motionProj[3][nrFilters][nrDirs] = {
    // Motion projection data...
};

#define scalingFiltSize 5
__constant__ float d_scalingFilt[scalingFiltSize] = {0.0884, 0.3536, 0.5303, 0.3536, 0.0884};

#define v1GaussFiltSize 9
__constant__ float d_v1GaussFilt[v1GaussFiltSize] = {0.0007, 0.0155, 0.0903, 0.2345, 0.3179, 0.2345, 0.0903, 0.0155, 0.0007};

#define complexV1FiltSize 11
__constant__ float d_complexV1Filt[complexV1FiltSize] = {0.0019, 0.0110, 0.0430, 0.1142, 0.2052, 0.2495, 0.2052, 0.1142, 0.0430, 0.0110, 0.0019};

#define normV1filtSize 25
__constant__ float d_normV1filt[normV1filtSize] = {0.0045, 0.0072, 0.0109, 0.0160, 0.0225, 0.0303, 0.0393, 0.0490, 0.0587, 0.0675, 0.0746, 0.0792, 0.0808, 0.0792, 0.0746, 0.0675, 0.0587, 0.0490, 0.0393, 0.0303, 0.0225, 0.0160, 0.0109, 0.0072, 0.0045};

#define diff1filtSize 3
__constant__ float d_diff1filt[diff1filtSize] = {-1/2.0, 0, 1/2.0};

#define diff2filtSize 3
__constant__ float d_diff2filt[diff2filtSize] = {1, -2, 1};

#define diff3filtSize 5
__constant__ float d_diff3filt[diff3filtSize] = {-1/2.0, 1, 0, -1, 1/2.0};

#define nrT 9

// Kernel for subtracting two images
__global__ void dev_sub(float *i1data, float *i2data, float* odata, int len) {
    const int tid = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
    const int threadN = IMUL(blockDim.x, gridDim.x);

    // Optimize with loop unrolling
    for(int i = tid; i < len; i += threadN) {
        odata[i] = i1data[i] - i2data[i];
    }
}