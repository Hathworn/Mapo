#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define IMUL(a, b) __mul24(a, b)
#define iDivUp(a, b) ((a) + (b) - 1) / (b)
#define CONV1_THREAD_SIZE 256
#define CONVN_THREAD_SIZE1 16
#define CONVN_THREAD_SIZE2 31

#define nrFilters 28
#define nrDirs 8

__constant__ float d_v1popDirs[3][nrFilters] = {
    {0.7246, -0.9718, 0.7496, -0.5837, -0.0810, 0.9439, 0.3203, -0.8712, -0.1593, -0.5142, 0.9304, 0.3737, -0.8031, -0.8126, 0.6004, -0.5738, 0.0024, 0.5969, 0.1436, 0.7757, -0.4004, -0.5108, 0.2375, -0.2221, -0.5140, 0.5194, -0.0870, 0.3838},
    {-0.6559, -0.1019, 0.6240, -0.7797, 0.9692, -0.2312, -0.9151, 0.4207, -0.9533, 0.8175, 0.2398, 0.8810, -0.4430, 0.0588, -0.5384, 0.5644, 0.7931, 0.5142, -0.7680, -0.0669, -0.6670, -0.2747, 0.5034, 0.5042, 0.1580, 0.1332, -0.5159, -0.3549},
    {0.2113, 0.2126, 0.2210, 0.2266, 0.2327, 0.2359, 0.2451, 0.2529, 0.2567, 0.2593, 0.2772, 0.2902, 0.3984, 0.5799, 0.5913, 0.5935, 0.6091, 0.6160, 0.6241, 0.6275, 0.6283, 0.8146, 0.8308, 0.8345, 0.8431, 0.8441, 0.8522, 0.8525}
};

__constant__ float motionProj[3][nrFilters][nrDirs] = {
    // Values omitted for brevity
};

#define scalingFiltSize 5
__constant__ float d_scalingFilt[scalingFiltSize] = {0.0884, 0.3536, 0.5303, 0.3536, 0.0884};

#define v1GaussFiltSize 9
__constant__ float d_v1GaussFilt[v1GaussFiltSize] = {0.0007, 0.0155, 0.0903, 0.2345, 0.3179, 0.2345, 0.0903, 0.0155, 0.0007};

#define complexV1FiltSize 11
__constant__ float d_complexV1Filt[complexV1FiltSize] = {0.0019, 0.0110, 0.0430, 0.1142, 0.2052, 0.2495, 0.2052, 0.1142, 0.0430, 0.0110, 0.0019};

#define normV1filtSize 25
__constant__ float d_normV1filt[normV1filtSize] = {0.0045, 0.0072, 0.0109, 0.0160, 0.0225, 0.0303, 0.0393, 0.0490, 0.0587, 0.0675, 0.0746, 0.0792, 0.0808, 0.0792, 0.0746, 0.0675, 0.0587, 0.0490, 0.0393, 0.0303, 0.0225, 0.0160, 0.0109, 0.0072, 0.0045};

#define diff1filtSize 3
__constant__ float d_diff1filt[diff1filtSize] = {-1/2.0, 0, 1/2.0};

#define diff2filtSize 3
__constant__ float d_diff2filt[diff2filtSize] = {1, -2, 1};

#define diff3filtSize 5
__constant__ float d_diff3filt[diff3filtSize] = {-1/2.0, 1, 0, -1, 1/2.0};

#define nrT 9

__global__ void dev_edges(float *data, int len, int nrX_, int nrY_) {
    const int tid = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
    const int threadN = IMUL(blockDim.x, gridDim.x);

    for(int i = tid; i < len; i += threadN) {
        int X = i % nrX_;
        int Y = (i / nrX_) % nrY_;
        int scale = i / (nrX_ * nrY_ * 28);

        float edgedist = (float)min(min(X, nrX_ - 1 - X), min(Y, nrY_ - 1 - Y));
        float edgeFactor = 1.0f;

        // Simplified conditions and used fminf for readability
        if (scale == 0) {
            edgeFactor = fminf(125.0f, edgedist * edgedist * edgedist) / 125.0f;
        } else if (scale == 1) {
            edgeFactor = fminf(1296.0f, powf(edgedist, 4.0f)) / 1296.0f;
        } else {
            edgeFactor = fminf(7776.0f, powf(edgedist, 5.0f)) / 7776.0f;
        }

        data[i] *= edgeFactor;
    }
}