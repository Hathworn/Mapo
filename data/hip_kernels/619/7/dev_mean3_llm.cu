#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define IMUL(a, b) __mul24(a, b)
#define iDivUp(a,b) ((a)+(b)-1)/(b)
#define CONV1_THREAD_SIZE 256
#define CONVN_THREAD_SIZE1 16
#define CONVN_THREAD_SIZE2 31 //31 is faster than 32 because shared memory is too full

#define nrFilters 28
#define nrDirs 8

__constant__ float d_v1popDirs[3][nrFilters] = {
{ 0.7246,-0.9718, 0.7496,-0.5837,-0.0810, 0.9439, 0.3203,-0.8712,-0.1593,-0.5142, 0.9304, 0.3737,-0.8031,-0.8126, 0.6004,-0.5738, 0.0024, 0.5969, 0.1436, 0.7757,-0.4004,-0.5108, 0.2375,-0.2221,-0.5140, 0.5194,-0.0870, 0.3838},
{-0.6559,-0.1019, 0.6240,-0.7797, 0.9692,-0.2312,-0.9151, 0.4207,-0.9533, 0.8175, 0.2398, 0.8810,-0.4430, 0.0588,-0.5384, 0.5644, 0.7931, 0.5142,-0.7680,-0.0669,-0.6670,-0.2747, 0.5034, 0.5042, 0.1580, 0.1332,-0.5159,-0.3549},
{ 0.2113, 0.2126, 0.2210, 0.2266, 0.2327, 0.2359, 0.2451, 0.2529, 0.2567, 0.2593, 0.2772, 0.2902, 0.3984, 0.5799, 0.5913, 0.5935, 0.6091, 0.6160, 0.6241, 0.6275, 0.6283, 0.8146, 0.8308, 0.8345, 0.8431, 0.8441, 0.8522, 0.8525}
};

__constant__ float motionProj[3][nrFilters][nrDirs] = {
// Data omitted for brevity
};

// Constants omitted for brevity

__global__ void dev_mean3(float *idata, float *odata, int nrXnrY, int nrZ) {
    int     tid = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
    int threadN = IMUL(blockDim.x, gridDim.x);
    int blockSize = nrXnrY * nrZ;

    for(int i = tid; i < nrXnrY; i += threadN) {
        // Use local register 'sum' instead of shared memory for efficiency
        float sum = 0.0f;
        int ind = i + blockIdx.y * blockSize;

        for (int j = 0; j < nrZ; j++) {
            // Ensure coalesced memory access by loading contiguous data
            sum += idata[ind + j * nrXnrY];
        }

        // Write the result directly to global memory
        odata[i + blockIdx.y * nrXnrY] = sum / nrZ;
    }
}