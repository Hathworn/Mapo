#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define IMUL(a, b) __mul24(a, b)
#define iDivUp(a,b) ((a)+(b)-1)/(b)
#define CONV1_THREAD_SIZE 256
#define CONVN_THREAD_SIZE1 16
#define CONVN_THREAD_SIZE2 31

#define nrFilters 28
#define nrDirs 8

__constant__ float d_v1popDirs[3][nrFilters] = {
    {0.7246, -0.9718, 0.7496, -0.5837, -0.0810, 0.9439, 0.3203, -0.8712, -0.1593, -0.5142, 0.9304, 0.3737, -0.8031, -0.8126, 0.6004, -0.5738, 0.0024, 0.5969, 0.1436, 0.7757, -0.4004, -0.5108, 0.2375, -0.2221, -0.5140, 0.5194, -0.0870, 0.3838},
    {-0.6559, -0.1019, 0.6240, -0.7797, 0.9692, -0.2312, -0.9151, 0.4207, -0.9533, 0.8175, 0.2398, 0.8810, -0.4430, 0.0588, -0.5384, 0.5644, 0.7931, 0.5142, -0.7680, -0.0669, -0.6670, -0.2747, 0.5034, 0.5042, 0.1580, 0.1332, -0.5159, -0.3549},
    {0.2113, 0.2126, 0.2210, 0.2266, 0.2327, 0.2359, 0.2451, 0.2529, 0.2567, 0.2593, 0.2772, 0.2902, 0.3984, 0.5799, 0.5913, 0.5935, 0.6091, 0.6160, 0.6241, 0.6275, 0.6283, 0.8146, 0.8308, 0.8345, 0.8431, 0.8441, 0.8522, 0.8525}
};

__constant__ float motionProj[3][nrFilters][nrDirs] = {
    // Initialization omitted for brevity.
};

#define scalingFiltSize 5
__constant__ float d_scalingFilt[scalingFiltSize] = {0.0884, 0.3536, 0.5303, 0.3536, 0.0884};

#define v1GaussFiltSize 9
__constant__ float d_v1GaussFilt[v1GaussFiltSize] = {0.0007, 0.0155, 0.0903, 0.2345, 0.3179, 0.2345, 0.0903, 0.0155, 0.0007};

#define complexV1FiltSize 11
__constant__ float d_complexV1Filt[complexV1FiltSize] = {0.0019, 0.0110, 0.0430, 0.1142, 0.2052, 0.2495, 0.2052, 0.1142, 0.0430, 0.0110, 0.0019};

#define normV1filtSize 25
__constant__ float d_normV1filt[normV1filtSize] = {0.0045, 0.0072, 0.0109, 0.0160, 0.0225, 0.0303, 0.0393, 0.0490, 0.0587, 0.0675, 0.0746, 0.0792, 0.0808, 0.0792, 0.0746, 0.0675, 0.0587, 0.0490, 0.0393, 0.0303, 0.0225, 0.0160, 0.0109, 0.0072, 0.0045};

#define diff1filtSize 3
__constant__ float d_diff1filt[diff1filtSize] = {-1/2.0, 0, 1/2.0};

#define diff2filtSize 3
__constant__ float d_diff2filt[diff2filtSize] = {1, -2, 1};

#define diff3filtSize 5
__constant__ float d_diff3filt[diff3filtSize] = {-1/2.0, 1, 0, -1, 1/2.0};

#define nrT 9

__global__ void dev_conv1(float* idata, float* odata, int len, const float* filt, int filtlen) {
    __shared__ float block[CONV1_THREAD_SIZE];

    // Calculate only necessary indices.
    const int nrValidConv = CONV1_THREAD_SIZE - (filtlen-1);
    const int offset = (filtlen-1)/2;

    int xInd = blockIdx.x * nrValidConv + threadIdx.x - offset;
    int idx = blockIdx.y * len + xInd;

    // Bounds check prior to loading into shared memory.
    block[threadIdx.x] = (xInd >= 0 && xInd < len) ? idata[idx] : 0;

    __syncthreads();

    xInd += offset;
    idx += offset;

    // Ensure index within valid compute bounds.
    if (xInd < len && threadIdx.x < nrValidConv) {
        float sum = 0;
        
        // Optimize loop unrolling manually for better performance.
        #pragma unroll
        for (int i = 0; i < filtlen; ++i) {
            sum += block[threadIdx.x + i] * filt[i];
        }
        
        odata[idx] = sum;
    }
}