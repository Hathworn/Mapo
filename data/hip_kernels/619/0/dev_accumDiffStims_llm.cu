#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define IMUL(a, b) __mul24(a, b)
#define iDivUp(a,b) ((a)+(b)-1)/(b)
#define CONV1_THREAD_SIZE 256
#define CONVN_THREAD_SIZE1 16
#define CONVN_THREAD_SIZE2 31 //31 is faster than 32 because shared memory is too full

// 28 space-time orientations of V1 simple cells
#define nrFilters 28

// 8 directions
#define nrDirs 8

__constant__ float d_v1popDirs[3][nrFilters] = {
{ 0.7246,-0.9718, 0.7496,-0.5837,-0.0810, 0.9439, 0.3203,-0.8712,-0.1593,-0.5142, 0.9304, 0.3737,-0.8031,-0.8126, 0.6004,-0.5738, 0.0024, 0.5969, 0.1436, 0.7757,-0.4004,-0.5108, 0.2375,-0.2221,-0.5140, 0.5194,-0.0870, 0.3838},
{-0.6559,-0.1019, 0.6240,-0.7797, 0.9692,-0.2312,-0.9151, 0.4207,-0.9533, 0.8175, 0.2398, 0.8810,-0.4430, 0.0588,-0.5384, 0.5644, 0.7931, 0.5142,-0.7680,-0.0669,-0.6670,-0.2747, 0.5034, 0.5042, 0.1580, 0.1332,-0.5159,-0.3549},
{ 0.2113, 0.2126, 0.2210, 0.2266, 0.2327, 0.2359, 0.2451, 0.2529, 0.2567, 0.2593, 0.2772, 0.2902, 0.3984, 0.5799, 0.5913, 0.5935, 0.6091, 0.6160, 0.6241, 0.6275, 0.6283, 0.8146, 0.8308, 0.8345, 0.8431, 0.8441, 0.8522, 0.8525}
};

extern __shared__ float motionProjDyn[];

__constant__ float motionProj[3][nrFilters][nrDirs] = { /* Data not shown for brevity */ };

#define scalingFiltSize 5
__constant__ float d_scalingFilt[scalingFiltSize] = {0.0884, 0.3536, 0.5303, 0.3536, 0.0884};

// d_v1GaussFilt defines the 1D receptive field size of a V1 unit
#define v1GaussFiltSize 9
__constant__ float d_v1GaussFilt[v1GaussFiltSize] = {0.0007, 0.0155, 0.0903, 0.2345, 0.3179, 0.2345, 0.0903, 0.0155, 0.0007};

// d_complexV1Filt is the spacial filter for complex cells
#define complexV1FiltSize 11
__constant__ float d_complexV1Filt[complexV1FiltSize] = {0.0019, 0.0110, 0.0430, 0.1142, 0.2052, 0.2495, 0.2052, 0.1142, 0.0430, 0.0110, 0.0019};

#define normV1filtSize 25
__constant__ float d_normV1filt[normV1filtSize]={0.0045,0.0072,0.0109,0.0160,0.0225,0.0303,0.0393,0.0490,0.0587,0.0675,0.0746,0.0792,0.0808,0.0792,0.0746,0.0675,0.0587,0.0490,0.0393,0.0303,0.0225,0.0160,0.0109,0.0072,0.0045};

#define diff1filtSize 3
__constant__ float d_diff1filt[diff1filtSize] = {-1/2.0, 0, 1/2.0};

#define diff2filtSize 3
__constant__ float d_diff2filt[diff2filtSize] = {1, -2, 1};

#define diff3filtSize 5
__constant__ float d_diff3filt[diff3filtSize] = {-1/2.0, 1, 0, -1, 1/2.0};

#define nrT 9

__global__ void dev_accumDiffStims(float *d_resp_tmp, float *diffV1GausBuf, int nrXnrY, int scale, int orderX, int orderY, int orderT) {
    const int tid = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
    const int threadN = IMUL(blockDim.x, gridDim.x);

    __shared__ float dirorders[nrFilters];

    if (threadIdx.x < nrFilters) {
        const float dir1 = d_v1popDirs[0][threadIdx.x]; // x-component
        const float dir2 = d_v1popDirs[1][threadIdx.x]; // y-component
        const float dir3 = d_v1popDirs[2][threadIdx.x]; // t-component

        float dirX = (orderX == 0) ? 1 : (orderX == 1) ? dir1 : (orderX == 2) ? dir1 * dir1 : dir1 * dir1 * dir1;
        float dirY = (orderY == 0) ? 1 : (orderY == 1) ? dir2 : (orderY == 2) ? dir2 * dir2 : dir2 * dir2 * dir2;
        float dirT = (orderT == 0) ? 1 : (orderT == 1) ? dir3 : (orderT == 2) ? dir3 * dir3 : dir3 * dir3 * dir3;
        dirorders[threadIdx.x] = dirX * dirY * dirT;
    }

    __syncthreads();

    // Optimize by reducing register pressure and using a single iteration variable
    for(int i = tid; i < nrXnrY * nrFilters; i += threadN) {
        int cell_idx = i / nrFilters; // Calculating index for memory access
        int filter_idx = i % nrFilters; // Calculating index for direction

        float d = diffV1GausBuf[cell_idx];
        d_resp_tmp[cell_idx + filter_idx * nrXnrY] += scale * d * dirorders[filter_idx];
    }
}