#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define IMUL(a, b) __mul24(a, b)
#define iDivUp(a,b) ((a)+(b)-1)/(b)
#define CONV1_THREAD_SIZE 256
#define CONVN_THREAD_SIZE1 16
#define CONVN_THREAD_SIZE2 31

#define nrFilters 28
#define nrDirs 8

__constant__ float d_v1popDirs[3][nrFilters] = {
{ 0.7246,-0.9718, 0.7496,-0.5837,-0.0810, 0.9439, 0.3203,-0.8712,-0.1593,-0.5142, 0.9304, 0.3737,-0.8031,-0.8126, 0.6004,-0.5738, 0.0024, 0.5969, 0.1436, 0.7757,-0.4004,-0.5108, 0.2375,-0.2221,-0.5140, 0.5194,-0.0870, 0.3838},
{-0.6559,-0.1019, 0.6240,-0.7797, 0.9692,-0.2312,-0.9151, 0.4207,-0.9533, 0.8175, 0.2398, 0.8810,-0.4430, 0.0588,-0.5384, 0.5644, 0.7931, 0.5142,-0.7680,-0.0669,-0.6670,-0.2747, 0.5034, 0.5042, 0.1580, 0.1332,-0.5159,-0.3549},
{ 0.2113, 0.2126, 0.2210, 0.2266, 0.2327, 0.2359, 0.2451, 0.2529, 0.2567, 0.2593, 0.2772, 0.2902, 0.3984, 0.5799, 0.5913, 0.5935, 0.6091, 0.6160, 0.6241, 0.6275, 0.6283, 0.8146, 0.8308, 0.8345, 0.8431, 0.8441, 0.8522, 0.8525}
};

// Optimized kernel to average two data arrays and store results
__global__ void dev_ave(float *i1data, float *i2data, float* odata, int len) {
    const int tid = IMUL(blockDim.x, blockIdx.x) + threadIdx.x;
    const int stride = IMUL(blockDim.x, gridDim.x);

    for (int i = tid; i < len; i += stride) {
        odata[i] = (i1data[i] + i2data[i]) * 0.5; // Use multiplication for better performance
    }
}