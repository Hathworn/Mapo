#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function
__global__ void SimpleClone(const float *background, const float *target, const float *mask, float *output, const int wb, const int hb, const int wt, const int ht, const int oy, const int ox)
{
    const int yt = blockIdx.y * blockDim.y + threadIdx.y;
    const int xt = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary and mask check combined to reduce conditional checks
    if (yt < ht && xt < wt) {
        const int curt = wt * yt + xt;
        if (mask[curt] > 127.0f) {
            const int yb = oy + yt, xb = ox + xt;
            if (yb < hb && xb < wb) {
                const int curb = wb * yb + xb;
                // Unrolling loop to improve instruction-level parallelism
                #pragma unroll
                for (int i = 0; i < 3; ++i) {
                    output[curb * 3 + i] = target[curt * 3 + i];
                }
            }
        }
    }
}