#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ZeroMeanImpl(float* solutions, int rowSize, int matCount) {
    const int matricesPerBlock = BLOCK_SIZE / rowSize;
    const int matrixIdx = blockIdx.x * matricesPerBlock + threadIdx.x / rowSize;
    const int tid = threadIdx.x;
    const int col = threadIdx.x % rowSize; // Use modulus for better readability
    const int inBlockOffset = threadIdx.x / rowSize;

    __shared__ double beta[BLOCK_SIZE];
    __shared__ double line[BLOCK_SIZE];

    if (matrixIdx >= matCount) {
        return;
    }

    solutions += matrixIdx * rowSize;
    beta[tid] = (col != (rowSize - 1)) ? solutions[col] : 0; // Conditional operator for clarity
    line[tid] = beta[tid];
    __syncthreads();

    // Efficient reduction with stride doubling
    for (int s = rowSize / 2; s > 0; s >>= 1) {
        if (col < s) {
            line[tid] += line[tid + s];
        }
        __syncthreads();
    }

    // Update solutions
    beta[tid] -= line[rowSize * inBlockOffset] / rowSize;
    solutions[col] = beta[tid];
}