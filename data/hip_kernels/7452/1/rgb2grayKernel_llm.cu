#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rgb2grayKernel(unsigned char *imgr, unsigned char *imgg, unsigned char *imgb, unsigned char *img_gray, int n) {

    // Optimize memory access by using local variables
    int index = blockIdx.x * blockDim.x + threadIdx.x;    

    if (index < n){
        unsigned char r = imgr[index];
        unsigned char g = imgg[index];
        unsigned char b = imgb[index];

        // Minimize calculation steps with precomputed constants
        img_gray[index] = (unsigned char)( 0.299f * r + 0.587f * g + 0.114f * b);
    }
}