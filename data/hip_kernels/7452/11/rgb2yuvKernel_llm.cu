#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rgb2yuvKernel(int *imgr, int *imgg, int *imgb, int *imgy, int *imgcb, int *imgcr, int n) {

    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Check if the thread index is within the array bounds
    if (index < n) {
        int r = imgr[index];
        int g = imgg[index];
        int b = imgb[index];

        // Compute Y, Cb, Cr using integer arithmetic for efficiency
        int y  = __float2int_rn(0.299f * r + 0.587f * g + 0.114f * b);
        int cb = __float2int_rn(-0.147f * r - 0.289f * g + 0.436f * b);
        int cr = __float2int_rn(0.615f * r - 0.515f * g - 0.1f * b);

        imgy[index] = y;
        imgcb[index] = cb;
        imgcr[index] = cr;
    }
}