#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void rgb2binaryKernel(unsigned char *imgr, unsigned char *imgg, unsigned char *imgb, unsigned char *img_binary, int n, int umbral) {
    // Use shared memory for local variables to reduce global memory traffic
    __shared__ float cached_r[256]; 
    __shared__ float cached_g[256]; 
    __shared__ float cached_b[256];

    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (index < n) {
        // Cache data in shared memory
        cached_r[threadIdx.x] = 0.299f * imgr[index];
        cached_g[threadIdx.x] = 0.587f * imgg[index];
        cached_b[threadIdx.x] = 0.114f * imgb[index];

        // Calculate binary image using cached values
        img_binary[index] = (cached_r[threadIdx.x] + cached_g[threadIdx.x] + cached_b[threadIdx.x] > umbral) ? 255 : 0;
    }
}