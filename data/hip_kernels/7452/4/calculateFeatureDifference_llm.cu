#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float computeS(float *sumTable, int rowNumberN, int colNumberM, int startX, int startY, int Kx, int Ky) {
    startX--;
    startY--;
    float S =
    sumTable[startX + Kx + (Ky + startY) * colNumberM] -
    (startX < 0 ? 0 : sumTable[startX + (Ky + startY) * colNumberM]) -
    (startY < 0 ? 0 : sumTable[startX + Kx + startY * colNumberM]) +
    (startX < 0 || startY < 0 ? 0 : sumTable[startX + startY * colNumberM]);
    return S;
}

__global__ void calculateFeatureDifference(float *templateFeatures, int colNumberM, int rowNumberN, float *l1SumTable, float *l2SumTable, float *lxSumTable, float *lySumTable, int Kx, int Ky, float *differences) {
    int widthLimit = colNumberM - Kx + 1;
    int heightLimit = rowNumberN - Ky + 1;

    int startX = threadIdx.x + blockIdx.x * blockDim.x;
    int startY = threadIdx.y + blockIdx.y * blockDim.y;

    // Return if the thread is out of bounds
    if (startX >= widthLimit || startY >= heightLimit) return;

    // Precompute reciprocal of Kx * Ky for reuse
    float reciprocalArea = 1.0f / (Kx * Ky);

    // Compute sums
    float S1D = computeS(l1SumTable, rowNumberN, colNumberM, startX, startY, Kx, Ky);
    float S2D = computeS(l2SumTable, rowNumberN, colNumberM, startX, startY, Kx, Ky);
    float SxD = computeS(lxSumTable, rowNumberN, colNumberM, startX, startY, Kx, Ky);
    float SyD = computeS(lySumTable, rowNumberN, colNumberM, startX, startY, Kx, Ky);

    // Compute mean and variance
    float meanVector = S1D * reciprocalArea;
    float varianceVector = (S2D * reciprocalArea) - powf(meanVector, 2);

    // Compute gradients
    float xCenter = startX + Kx / 2.0f;
    float yCenter = startY + Ky / 2.0f;
    float xGradientVector = 4 * (SxD - xCenter * S1D) / (Kx * Kx * Ky);
    float yGradientVector = 4 * (SyD - yCenter * S1D) / (Ky * Ky * Kx);

    // Store differences using norm4df
    differences[startX + startY * widthLimit] = norm4df(
        templateFeatures[0] - meanVector, 
        templateFeatures[1] - varianceVector,
        templateFeatures[2] - xGradientVector,
        templateFeatures[3] - yGradientVector
    );
}