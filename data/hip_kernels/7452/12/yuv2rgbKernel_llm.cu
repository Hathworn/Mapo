#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yuv2rgbKernel(int *imgy, int *imgcb, int *imgcr, int *imgr, int *imgg, int *imgb, int n) {

    // Calculate thread index
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // Check if the thread index is within bounds
    if (index < n) {
        // Load YUV values
        int y = imgy[index];
        int cb = imgcb[index];
        int cr = imgcr[index];

        // Compute RGB values using integer arithmetic
        int r = y + (int)(1.14f * cr);
        int g = y - (int)(0.396f * cb + 0.581f * cr);
        int b = y + (int)(2.029f * cb);

        // Store results
        imgr[index] = r;
        imgg[index] = g;
        imgb[index] = b;
    }
}