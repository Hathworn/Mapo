#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calcSumTable(const float *rowCumSum, float *SumTable, int rowNumberN, int colNumberM) {
    int xIndex = threadIdx.x + blockIdx.x * blockDim.x;
    if (xIndex >= colNumberM) return;

    // Use shared memory to reduce global memory accesses
    extern __shared__ float sharedMem[];
    
    for (int i = 1; i < rowNumberN; i++) {
        int globalIndex = i * colNumberM + xIndex;
        int prevGlobalIndex = (i - 1) * colNumberM + xIndex;

        // Load data into shared memory
        sharedMem[threadIdx.x] = rowCumSum[prevGlobalIndex];
        __syncthreads();

        // Update SumTable with values from shared memory
        SumTable[globalIndex] += sharedMem[threadIdx.x];
        __syncthreads(); // Ensure all threads have completed their updates
    }
}