#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_add(float* a, float* b, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use loop unrolling for better performance
    int stride = gridDim.x * blockDim.x;
    for (; i < n; i += stride) {
        a[i] = a[i] + b[i];
    }
}