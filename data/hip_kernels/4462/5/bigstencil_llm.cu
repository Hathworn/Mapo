#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using shared memory for coalescing reads
__global__ void bigstencil(int* in, int* out) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Allocate shared memory
    extern __shared__ int shared_in[];

    // Load data into shared memory
    shared_in[threadIdx.x] = in[i];
    __syncthreads();

    // Perform computation using shared memory and write to global memory
    out[i] = shared_in[threadIdx.x] + 2;
}