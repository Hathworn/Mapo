#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_multiply_kernel(double *matrix, double *vector_in, double *vector_out, long dim_mn) {
    // Pre-compute indices and check bounds
    long i = threadIdx.x + blockIdx.x * blockDim.x;

    if (i < dim_mn) {
        double out = 0.0;

        // Use shared memory to minimize access latency
        __shared__ double local_vector_in[1024];
        
        long threadIdx_x = threadIdx.x;
        
        // Load vector_in to shared memory
        if (threadIdx_x < dim_mn) {
            local_vector_in[threadIdx_x] = vector_in[threadIdx_x];
        }
        __syncthreads();
        
        // Reduced access into vector_in by using local_vector_in
        for (long j = 0; j < dim_mn; j++) {
            out += matrix[i * dim_mn + j] * local_vector_in[j];
        }

        vector_out[i] = out;
    }
}