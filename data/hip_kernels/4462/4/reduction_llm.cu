#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduction(int* input, int* output) {
    __shared__ int tmp[TPB];

    // Load input into shared memory
    int tid = threadIdx.x;
    int index = tid + blockIdx.x * blockDim.x;
    tmp[tid] = input[index];
    __syncthreads();

    // Block-wide reduction using a loop
    // Unroll loop for further optimization
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            tmp[tid] += tmp[tid + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = tmp[0];
    }
}