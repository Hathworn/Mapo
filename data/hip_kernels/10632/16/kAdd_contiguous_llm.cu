#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int stride = blockDim.x * gridDim.x;
    
    // Use for-loop with stride for better memory access pattern
    for (unsigned int i = idx; i < numEls; i += stride) {
        dest[i] = a[i] + b[i];
    }
}