#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

__device__ int getPosition(int x, int y, int width, int margin, int pixelPosition)
{
    return (x + (y * width)) * margin + pixelPosition;
}

__global__ void addEffect(unsigned char* output_img, unsigned char* input_img, int width, int height, int nbBlocks)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global x index
    int y = blockIdx.y * blockDim.y + threadIdx.y; // Calculate global y index

    if (x < width && y < height) {
        int currentIndex = getPosition(x, y, width, 3, 0);
        if ((input_img[currentIndex] + input_img[currentIndex + 1] + input_img[currentIndex + 2]) / 3 < 20) {
            output_img[currentIndex] = input_img[currentIndex];
            output_img[currentIndex + 1] = input_img[currentIndex + 1];
            output_img[currentIndex + 2] = input_img[currentIndex + 2];

            for (int i = -4; i <= 4; i++) {
                for (int j = -4; j <= 4; j++) {
                    int nx = x + i;
                    int ny = y + j;
                    if (nx >= 0 && nx < width && ny >= 0 && ny < height) {
                        int neighbourIndex = getPosition(nx, ny, width, 3, 0);
                        output_img[neighbourIndex] = 0;
                        output_img[neighbourIndex + 1] = 0;
                        output_img[neighbourIndex + 2] = 0;
                    }
                }
            }
        }
    }
}