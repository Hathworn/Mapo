#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setGroupsPointersDead(multipassConfig_t* mbk, unsigned numBuckets)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Use block and thread indices for better parallelism
    if(index < numBuckets)
    {
        mbk->isNextDeads[index] = 1; // Direct assignment without unnecessary operations
    }
}