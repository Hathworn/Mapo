#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setGroupsPointersDead(multipassConfig_t* mbk, unsigned numBuckets)
{
    // Use blockIdx and blockDim for parallel execution
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < numBuckets)
    {
        mbk->isNextDeads[index] = 1;
    }
}