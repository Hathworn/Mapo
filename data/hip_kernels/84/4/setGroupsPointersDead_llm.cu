#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setGroupsPointersDead(multipassConfig_t* mbk, unsigned numBuckets)
{
    // Use shared memory and avoid accessing global memory inside the loop
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Use block and thread indices for 1D grid

    if(index < numBuckets)
    {
        mbk->isNextDeads[index] = 1; // Efficiently setting the value directly
    }
}