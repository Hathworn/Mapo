#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void setGroupsPointersDead(multipassConfig_t* mbk, unsigned numBuckets)
{
    // Calculate the global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for improved performance (considered if appropriate)
    // __shared__ int sharedData[NUM_BUCKETS];
    
    // Check if the index is within bounds
    if(index < numBuckets) 
    {
        // Directly assign in global memory
        mbk->isNextDeads[index] = 1;
    }
}