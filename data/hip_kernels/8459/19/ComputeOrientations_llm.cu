#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOrientations(float *g_Data, int *d_Ptrs, float *d_Orient, int maxPts, int w, int h)
{
    // Use smaller shared memory allocation
    __shared__ float data[16*15];
    __shared__ float hist[32];
    __shared__ float gauss[16];
    
    const int tx = threadIdx.x;
    const int bx = blockIdx.x;
    
    if (tx < 32) {
        hist[tx] = 0.0f; // Initialize histogram with zeros
    }
    __syncthreads();
    
    float i2sigma2 = -1.0f/(2.0f*3.0f*3.0f);
    if (tx < 15) {
        gauss[tx] = exp(i2sigma2*(tx-7)*(tx-7)); // Precompute Gaussian weights
    }
    
    int p = d_Ptrs[bx];
    int yp = p / w - 7;
    int xp = p % w - 7;
    int px = xp & 15;
    int x = tx - px;

    // Load the data into shared memory
    for (int y = 0; y < 15; y++) {
        int memPos = 16 * y + x;
        int xi = xp + x;
        int yi = yp + y;

        xi = max(0, min(w-1, xi)); // Clamp the coordinates
        yi = max(0, min(h-1, yi)); // Clamp the coordinates

        if (x >= 0 && x < 15) {
            data[memPos] = g_Data[yi * w + xi];
        }
    }
    __syncthreads();
    
    // Compute gradients and update histograms
    for (int y = 1; y < 14; y++) {
        int memPos = 16 * y + x;
        if (x >= 1 && x < 14) {
            float dy = data[memPos + 16] - data[memPos - 16];
            float dx = data[memPos + 1] - data[memPos - 1];
            int bin = roundf(16.0f * atan2f(dy, dx) / 3.1416f + 16.5f);
            bin = bin == 32 ? 0 : bin;
            float grad = sqrtf(dx * dx + dy * dy);
            atomicAdd(&hist[bin], grad * gauss[x] * gauss[y]); // Atomic operation to prevent race condition
        }
    }
    __syncthreads();
    
    // Find peaks in the histogram
    float v = hist[tx];
    hist[tx] = (v > hist[(tx + 1) & 31] && v >= hist[(tx + 31) & 31] ? v : 0.0f);
    __syncthreads();
    
    if (tx == 0) {
        float maxval1 = 0.0f;
        float maxval2 = 0.0f;
        int i1 = -1;
        int i2 = -1;
        
        for (int i = 0; i < 32; i++) {
            float v = hist[i];
            if (v > maxval1) {
                maxval2 = maxval1;
                maxval1 = v;
                i2 = i1;
                i1 = i;
            } else if (v > maxval2) {
                maxval2 = v;
                i2 = i;
            }
        }

        // Compute subpixel peak
        float val1 = hist[(i1 + 1) & 31];
        float val2 = hist[(i1 + 31) & 31];
        float peak = i1 + 0.5f * (val1 - val2) / (2.0f * maxval1 - val1 - val2);
        d_Orient[bx] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        
        if (maxval2 < 0.8f * maxval1) {
            i2 = -1;
        }
        
        if (i2 >= 0) {
            val1 = hist[(i2 + 1) & 31];
            val2 = hist[(i2 + 31) & 31];
            float peak = i2 + 0.5f * (val1 - val2) / (2.0f * maxval2 - val1 - val2);
            d_Orient[bx + maxPts] = 11.25f * (peak < 0.0f ? peak + 32.0f : peak);
        } else {
            d_Orient[bx + maxPts] = i2;
        }
    }
}