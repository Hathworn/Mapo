#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x for dynamic block width
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Use blockDim.y for dynamic block height
    int p = y * pitch + x;
    
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p]; // Perform subtraction if within bounds
    }
}
```
