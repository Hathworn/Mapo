#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void Subtract(float *d_Result, float *d_Data1, float *d_Data2, int width, int height)
{
    // Calculate coordinates using block and thread index
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int p = y * width + x;

    // Check if within bounds and perform subtraction
    if (x < width && y < height) {
        d_Result[p] = d_Data1[p] - d_Data2[p];
    }
}