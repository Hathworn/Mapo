#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MultiplyAdd(float *d_Result, float *d_Data, int width, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified thread ID calculation
    const int y = blockIdx.y * blockDim.y + threadIdx.y;  // Simplified thread ID calculation
    if (x < width && y < height)
    {
        int p = y * width + x;  // Updated position calculation
        d_Result[p] = d_ConstantA[0] * d_Data[p] + d_ConstantB[0];
    }
}