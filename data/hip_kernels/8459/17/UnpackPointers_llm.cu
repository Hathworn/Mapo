#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    // Calculate global thread index
    const int tx = threadIdx.x + blockIdx.x * blockDim.x;
    int numPts = 0;

    // Iterate over chunks of rows
    for (int y = 0; y < h / 32; y++) {
        for (int x = 0; x < w; x += 16) {
            // Ensure accessing within bounds
            if (x + tx < w) {
                unsigned int val = minmax[y * w + x + tx];
                if (val) {
                    for (int k = 0; k < 32; k++) {
                        if ((val & 0x1) && (numPts < maxPts)) {
                            // Calculate and store the pointer
                            ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                            numPts++;
                        }
                        val >>= 1;
                    }
                }
            }
        }
    }
}