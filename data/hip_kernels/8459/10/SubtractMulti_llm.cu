#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void SubtractMulti(float *d_Result, float *d_Data, int width, int pitch, int height)
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x; // Use blockDim.x for improved flexibility
    const int y = blockIdx.y * blockDim.y + threadIdx.y; // Use blockDim.y for improved flexibility
    const int sz = height * pitch;
    const int p = threadIdx.z * sz + y * pitch + x;
    
    if (x < width && y < height) {
        d_Result[p] = d_Data[p] - d_Data[p + sz];
    }
    // Remove __syncthreads(); as it is not required here
}