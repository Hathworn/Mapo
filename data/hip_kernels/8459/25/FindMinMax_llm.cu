#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void FindMinMax(float *d_MinMax, float *d_Data, int width, int height)
{
    __shared__ float minvals[128];
    __shared__ float maxvals[128];
    const int tx = threadIdx.x;
    const int x = blockIdx.x * 128 + tx;
    const int y = blockIdx.y * 16;
    const int b = blockDim.x;
    int p = y * width + x;
    
    // Ensure threads work on valid data
    if (x < width) {
        float val = d_Data[p];
        minvals[tx] = val;
        maxvals[tx] = val;
        // Iterate over the rows within the block
        for (int ty = 1; ty < 16; ty++) {
            p += width;
            val = d_Data[p];
            if (val < minvals[tx])
                minvals[tx] = val;
            if (val > maxvals[tx])
                maxvals[tx] = val;
        }
    } else {
        float dummyVal = d_Data[p - x];  // Use dummy value for threads outside boundary
        minvals[tx] = dummyVal;
        maxvals[tx] = dummyVal;
    }

    __syncthreads();
    
    // Perform reduction to obtain block-wise min/max
    for (int d = b / 2; d >= 1; d >>= 1) {
        if (tx < d) {
            minvals[tx] = fminf(minvals[tx], minvals[tx + d]);
            maxvals[tx] = fmaxf(maxvals[tx], maxvals[tx + d]);
        }
        __syncthreads();
    }
    
    // Write block results to global memory
    if (tx == 0) {
        int ptr = 2 * (gridDim.x * blockIdx.y + blockIdx.x);
        d_MinMax[ptr] = minvals[0];
        d_MinMax[ptr + 1] = maxvals[0];
    }
    __syncthreads();
}