#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void UnpackPointers(int *minmax, int *ptrs, int w, int h, int maxPts)
{
    const int tx = threadIdx.x;
    int numPts = 0;
    // Use shared memory for better performance
    __shared__ unsigned int sharedMinMax[blockDim.x];
    for (int y = 0; y < h / 32; y++) {
        for (int x = 0; x < w; x += 16) {
            // Load minmax data into shared memory
            sharedMinMax[tx] = minmax[y * w + x + tx];
            __syncthreads(); // Ensure all threads have loaded minmax data
            
            unsigned int val = sharedMinMax[tx];
            if (val) {
                for (int k = 0; k < 32; k++) {
                    if (val & 0x1 && numPts < maxPts) {
                        ptrs[16 * numPts + tx] = (y * 32 + k) * w + x + tx;
                        numPts++;
                    }
                    val >>= 1;
                }
            }
            __syncthreads(); // Ensure all threads complete before next iteration
        }
    }
}