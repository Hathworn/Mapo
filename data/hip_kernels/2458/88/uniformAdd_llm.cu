#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    __shared__ float uni;
    // Load uniform value only once per block
    if (threadIdx.x == 0) {
        uni = uniforms[blockIdx.x + blockOffset];
    }
    
    unsigned int address = blockIdx.x * (blockDim.x * 2) + baseIndex + threadIdx.x;

    __syncthreads(); // Ensure uni is loaded before use

    // Perform two additions per thread with bounds check for second addition
    if (address < n) {
        g_data[address] += uni;
    }
    if (address + blockDim.x < n) {
        g_data[address + blockDim.x] += uni;
    }
}