#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Calculate bottom left corner position
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + j];

    // Function to perform atomic operations, reducing redundant code
    auto atomicOperation = [&w, &h, &image_stride, &dst, &normalization_factor](int tx, int ty, float weight, float value) {
        if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
            _atomicAdd(dst + ty * image_stride + tx, value * weight);
            _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
        }
    };

    // Fill pixel containing bottom right corner
    atomicOperation(tx, ty, dx * dy, value);

    // Fill pixel containing bottom left corner
    atomicOperation(tx - 1, ty, (1.0f - dx) * dy, value);

    // Fill pixel containing upper left corner
    atomicOperation(tx - 1, ty - 1, (1.0f - dx) * (1.0f - dy), value);

    // Fill pixel containing upper right corner
    atomicOperation(tx, ty - 1, dx * (1.0f - dy), value);
}