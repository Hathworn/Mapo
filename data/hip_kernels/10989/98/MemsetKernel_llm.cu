#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate the global index for the thread
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    // Check if the thread is within image bounds
    if (i < h && j < w) {
        // Compute 1D index from 2D coordinates
        const int pos = i * w + j;
        // Set image value at the computed index
        image[pos] = value;
    }
}