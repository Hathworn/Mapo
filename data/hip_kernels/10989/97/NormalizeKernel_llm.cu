#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate the global thread index directly to reduce the number of arithmetic operations
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Quickly exit if the thread is out of bounds
    if (i >= h || j >= w) return;

    const int pos = i * s + j;

    // Use a temporary variable to store the scale value to reduce memory access
    float scale = normalization_factor[pos];
    float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

    // Perform the scaling operation
    image[pos] *= invScale;
}