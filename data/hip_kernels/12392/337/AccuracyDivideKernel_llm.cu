#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load accuracy to a register for faster access
    float acc = *accuracy;

    // Perform division
    acc /= N;

    // Store result back to global memory
    *accuracy = acc;
}