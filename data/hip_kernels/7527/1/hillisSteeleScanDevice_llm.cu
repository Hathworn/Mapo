#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hillisSteeleScanDevice(int *d_array, int numberOfElements, int *d_tmpArray, int moveIndex)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;

    // Ensure index is within bounds
    if (index < numberOfElements && index >= moveIndex)
    {
        // Perform in-place update to improve memory access patterns
        d_tmpArray[index] += d_array[index - moveIndex];
    }
}