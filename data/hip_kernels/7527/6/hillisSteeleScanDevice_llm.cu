#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hillisSteeleScanDevice(int *d_array , int numberOfElements, int *d_tmpArray, int moveIndex) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    
    // Check within bounds
    if(index < numberOfElements) {
        d_tmpArray[index] = d_array[index];
        
        // Only perform addition if valid index
        if(index >= moveIndex) {
            d_tmpArray[index] += d_array[index - moveIndex];
        }
    }
}