#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function with optimized thread index access
__global__ void print() 
{
    // Using block index and block dimension to uniquely identify each thread globally
    int globalId = blockDim.x * blockIdx.x + threadIdx.x;
    printf("hello from gpu thread %d\n", globalId);
}