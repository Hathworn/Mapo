#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void serialReduction(int *d_array, int numberOfElements)
{
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to store intermediate sums
    extern __shared__ int s_data[];

    // Load elements into shared memory and perform reduction in parallel
    int sum = 0;
    for(int i = tid; i < numberOfElements; i += blockDim.x * gridDim.x)
    {
        sum += d_array[i];
    }
    s_data[threadIdx.x] = sum;
    __syncthreads();

    // Perform reduction within the block
    for(int stride = blockDim.x / 2; stride > 0; stride /= 2)
    {
        if(threadIdx.x < stride)
        {
            s_data[threadIdx.x] += s_data[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Thread 0 of each block outputs the block result
    if(threadIdx.x == 0)
    {
        atomicAdd(d_array, s_data[0]);
    }
}