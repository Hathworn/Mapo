#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scatter(int *d_array, int *d_predicateArray, int *d_scanArray, int *d_compactedArray, int d_numberOfElements)
{
    // Calculate global thread index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within valid range
    if (index < d_numberOfElements)
    {
        // Check if the element should be compacted
        if (d_predicateArray[index] == 1)
        {
            // Pre-fetch the target index for compaction
            int targetIndex = d_scanArray[index] - 1;
            
            // Perform the scatter operation efficiently
            d_compactedArray[targetIndex] = d_array[index];
        }
    }
}