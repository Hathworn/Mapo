#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void getPos(const int * __restrict__ d_scanArray, const int d_numberOfElements, int * __restrict__ d_lastPos)
{
    // Use shared memory for reduced latency access
    extern __shared__ int sharedMem[];
    
    // Only one thread accesses the last position
    if (threadIdx.x == 0) {
        // Read the last element into shared memory
        sharedMem[0] = d_scanArray[d_numberOfElements - 1];
    }
    __syncthreads();
    
    // Write the value from shared memory to the output
    if (threadIdx.x == 0) {
        *d_lastPos = sharedMem[0];
    }
}