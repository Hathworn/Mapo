#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hillisSteeleScanDevice(int *d_predicateArray, int d_numberOfElements, int *d_tmpArray, int d_offset)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < d_numberOfElements) 
    {
        // Direct access to prevent re-fetching and reduce memory reads
        int currentVal = d_predicateArray[index];
        d_tmpArray[index] = currentVal;
        
        // Use shared memory for temporary storage, reduces access time
        __shared__ int sharedData[BLOCK_SIZE];
        sharedData[threadIdx.x] = currentVal;
        __syncthreads();

        if (index - d_offset >= 0) 
        {
            // Improved memory access patterns
            int offsetVal = (threadIdx.x >= d_offset) ? sharedData[threadIdx.x - d_offset] : d_predicateArray[index - d_offset];
            d_tmpArray[index] = currentVal + offsetVal;
        }
    }
}