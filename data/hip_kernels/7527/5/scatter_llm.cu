#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scatter(int *d_array, int *d_scanArray, int *d_predicateArray, int *d_scatteredArray, int d_numberOfElements, int offset)
{
    // Calculate the global index for the current thread
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the index is within bounds
    if (index < d_numberOfElements)
    {
        int predicateValue = d_predicateArray[index];

        // Only scatter if predicateArray value is 1
        if (predicateValue == 1)
        {
            int scatterIndex = d_scanArray[index] - 1 + offset;

            // Place the value in the correct position in the scattered array
            d_scatteredArray[scatterIndex] = d_array[index];
        }
    }
}