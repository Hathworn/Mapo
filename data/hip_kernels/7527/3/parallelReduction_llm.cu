#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void parallelReduction(int *d_array, int numberOfElements, int elementsPerThread, int numberOfThreadsPerBlock, int numberOfBlocks, int *d_global)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    index = index * elementsPerThread;

    if (index >= numberOfElements) // Check for valid index
    {
        return;
    }

    int sum = 0;
    for (int i = index; i < index + elementsPerThread && i < numberOfElements; i++) // Avoid out-of-bounds access
    {
        sum += d_array[i];
    }

    extern __shared__ int d_blockMemmory[];
    d_blockMemmory[threadIdx.x] = sum;
    __syncthreads();

    // Use a reduction pattern to optimize block-level summing
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (threadIdx.x < s)
        {
            d_blockMemmory[threadIdx.x] += d_blockMemmory[threadIdx.x + s];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) // Write result of each block to global memory
    {
        d_global[blockIdx.x] = d_blockMemmory[0];
    }
}