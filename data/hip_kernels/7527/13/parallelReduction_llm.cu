#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void parallelReduction(int *d_array, int numberOfElements, int elementsPerThread, int numberOfThreadsPerBlock, int numberOfBlocks, int *d_global)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    // Compute local sum
    for (int i = index; i < numberOfElements; i += numberOfBlocks * numberOfThreadsPerBlock)
    {
        sum += d_array[i];
    }

    // Use shared memory for block-level reduction
    extern __shared__ int d_blockMemmory[];
    d_blockMemmory[threadIdx.x] = sum;
    __syncthreads();

    // Perform binary reduction to minimize serial summation
    for (unsigned int stride = numberOfThreadsPerBlock / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride)
        {
            d_blockMemmory[threadIdx.x] += d_blockMemmory[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Store the result in global memory
    if (threadIdx.x == 0)
    {
        d_global[blockIdx.x] = d_blockMemmory[0];
    }
}