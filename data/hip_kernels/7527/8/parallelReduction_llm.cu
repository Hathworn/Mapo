#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void parallelReduction(int *d_array, int numberOfElements, int elementsPerThread, int numberOfThreadsPerBlock, int numberOfBlocks, int *d_global)
{
    __shared__ int sharedSum[1024]; // Shared memory for intra-block reduction
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int localSum = 0;

    // Process elements in a strided manner
    for (int i = index; i < numberOfElements; i += gridDim.x * blockDim.x)
    {
        localSum += d_array[i];
    }

    sharedSum[threadIdx.x] = localSum; // Store local sum in shared memory
    __syncthreads();

    // Intra-block reduction using shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (threadIdx.x < stride)
        {
            sharedSum[threadIdx.x] += sharedSum[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write result for each block's reduction to global memory
    if (threadIdx.x == 0)
    {
        d_global[blockIdx.x] = sharedSum[0];
    }
}