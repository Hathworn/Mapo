#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void predicateDevice(int *d_array, int *d_predicateArrry, int d_numberOfElements, int bit, int bitset)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < d_numberOfElements)
    {
        // Use ternary operator for concise conditionals
        d_predicateArrry[index] = ((d_array[index] & bit) == 0) ? (bitset == 0) : (bitset != 0);
    }
}