#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multMatriz(float *da, float *db, float *dc, int num) {
    // Calculate row and column index for this thread
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    int i = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure index is within bounds
    if (i < num && j < num) {
        float sum = 0.0f;
        // Perform matrix multiplication for this thread's row and column
        for (unsigned int k = 0; k < num; k++) {
            sum += da[i * num + k] * db[k * num + j];
        }
        // Store the result in the output matrix
        dc[i * num + j] = sum;
    }
}