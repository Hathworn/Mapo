#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void MatrixMulSh( float *Md , float *Nd , float *Pd , const int WIDTH )
{
    // Use shared memory for sub-matrices from Md and Nd
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // Calculate global column and row for current thread
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

    // Initialize the accumulator for the output element
    float Pvalue = 0;

    // Loop over all the tiles required to compute the block sub-matrix
    for (int m = 0; m < WIDTH / TILE_WIDTH; ++m)
    {
        // Load Md and Nd tiles into shared memory
        Mds[threadIdx.y][threadIdx.x] = Md[row * WIDTH + (m * TILE_WIDTH + threadIdx.x)];
        Nds[threadIdx.y][threadIdx.x] = Nd[(m * TILE_WIDTH + threadIdx.y) * WIDTH + col];

        // Synchronize to make sure the tiles are loaded
        __syncthreads();

        // Perform the multiplication for the tile
        for (int k = 0; k < TILE_WIDTH; ++k)
        {
            Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        }

        // Synchronize to make sure that the preceding computation is done before loading new tile into shared memory
        __syncthreads();
    }

    // Write the block sub-matrix to global memory
    Pd[row * WIDTH + col] = Pvalue;
}