#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void indices() {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    double sqrt_val = sqrt((double)id); // Precompute sqrt to avoid recalculation
    printf("blockdimy: %d  threadx: %d  Blockidx: %d  blockdimx: %d id:  %d raiz: %f\n",
           blockDim.y, threadIdx.x, blockIdx.x, blockDim.x, id, sqrt_val);
    // Removed __syncthreads(); as it is unnecessary in this context
}