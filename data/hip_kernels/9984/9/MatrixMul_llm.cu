#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMul(float *Md, float *Nd, float *Pd, const int WIDTH) {
    // Calculate thread id
    unsigned int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    unsigned int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    
    // Ensure indices are within bounds
    if (row < WIDTH && col < WIDTH) {
        float Pvalue = 0;
        for (int k = 0; k < WIDTH; ++k) {
            Pvalue += Md[row * WIDTH + k] * Nd[k * WIDTH + col];
        }
        Pd[row * WIDTH + col] = Pvalue; // Store result in Pd
    }
}