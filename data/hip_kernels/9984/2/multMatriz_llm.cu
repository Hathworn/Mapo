#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multMatriz(float *da, float *db, float *dc, int num) {
    // Use shared memory for faster data access
    __shared__ float tileA[32][32];
    __shared__ float tileB[32][32];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0;

    // Loop over tiles
    for (int m = 0; m < (num + blockDim.x - 1) / blockDim.x; ++m) {
        // Load data into shared memory
        if (row < num && m * blockDim.x + threadIdx.x < num) 
            tileA[threadIdx.y][threadIdx.x] = da[row * num + m * blockDim.x + threadIdx.x];
        else 
            tileA[threadIdx.y][threadIdx.x] = 0.0f;

        if (col < num && m * blockDim.y + threadIdx.y < num) 
            tileB[threadIdx.y][threadIdx.x] = db[(m * blockDim.y + threadIdx.y) * num + col];
        else 
            tileB[threadIdx.y][threadIdx.x] = 0.0f;

        // Wait for all threads to load their data
        __syncthreads();

        // Compute partial sums
        for (int e = 0; e < blockDim.x; ++e) {
            sum += tileA[threadIdx.y][e] * tileB[e][threadIdx.x];
        }

        // Wait for all threads to complete computation
        __syncthreads();
    }

    // Write result to global memory
    if (row < num && col < num) {
        dc[row * num + col] = sum;
    }
}