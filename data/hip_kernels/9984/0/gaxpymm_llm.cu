#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpymm(double *y, double *a, double *b, int m, int n, int p) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];

    // Ensure valid block index and thread index
    if (bid < m && tid < n) {
        for (int c = 0; c < p; c++) {
            // Load multiplication results to shared memory
            dots_s[tid * p + c] = a[bid * n + tid] * b[tid * p + c];
        }
        __syncthreads(); // Synchronize threads to ensure shared memory is ready

        // Accumulate results using a single thread
        if (tid == 0) {
            for (int c = 0; c < p; c++) {
                double sum = dots_s[c];
                for (int i = 1; i < n; i++) {
                    sum += dots_s[i * p + c];
                }
                y[bid * p + c] = sum; // Write results to global memory
            }
        }
    }
}