#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void suma(int a, int b, int *c) {
    // Utilize shared memory to speed up computation
    __shared__ int temp; 
    
    // Ensure the operation is only performed by one thread
    if (threadIdx.x == 0) {
        temp = a + b;
    }
    
    // Use __syncthreads to ensure all threads have the result
    __syncthreads(); 
    
    if (threadIdx.x == 0) {
        *c = temp;
    }
}