#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpy(double *y, double *a, double *x, int m, int n) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    double sum = 0.0;  // Accumulator for partial sums

    if (bid < m) {
        for (int i = tid; i < n; i += blockDim.x) {
            sum += a[bid * n + i] * x[i];  // Accumulate products
        }
        // Use atomic operation to sum partial results
        atomicAdd(&y[bid], sum);
    }
}