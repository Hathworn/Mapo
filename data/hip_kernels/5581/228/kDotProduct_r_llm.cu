#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];
    
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Loop unrolling for efficiency
    if (eidx < numElements) {
        for (uint stride = gridDim.x * DP_BLOCKSIZE; eidx < numElements; eidx += stride) {
            sum += a[eidx] * b[eidx];
        }
    }
    
    shmem[threadIdx.x] = sum;
    __syncthreads();

    // Optimize reduction using loop unrolling
    for (uint stride = DP_BLOCKSIZE / 2; stride > 32; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }
    
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}