#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Unique index for each thread
    const int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    
    // Loop through target elements with stride
    for (uint i = index; i < tgtWidth * tgtHeight; i += stride) {
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        tgt[i] = src[srcY * srcWidth + srcX]; // Copy from source to target
    }
}