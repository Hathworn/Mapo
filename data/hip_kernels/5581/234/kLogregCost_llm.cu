#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;
    
    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];
        
        labelLogProbs[tx] = __logf(labelp);
        
        // Compute probability of selecting the most probable label
        if (labelp != maxp) {
            correctProbs[tx] = 0.0f;
        } else {
            int numMax = 0;
            float maxProbVal = 0.0f;
            for (int i = 0; i < numOut; i++) {
                // Unroll loop for efficiency and leverage for parallel comparison
                float currentProb = probs[i * numCases + tx];
                if (currentProb == maxp) {
                    numMax++;
                    maxProbVal = currentProb;
                }
            }
            correctProbs[tx] = 1.0f / float(numMax);
        }
    }
}