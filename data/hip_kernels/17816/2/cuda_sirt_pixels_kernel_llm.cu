#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_sirt_pixels_kernel(int p, int nx, int dx, float* recon, const float* data)
{
    int d0 = blockIdx.x * blockDim.x + threadIdx.x;
    int dstride = blockDim.x * gridDim.x;

    for (int d = d0; d < dx; d += dstride)
    {
        float sum = 0.0f;

        // Use shared memory for partial sums to reduce global memory access
        __shared__ float partialSum[256]; // assuming blockDim.x <= 256
        int tid = threadIdx.x;
        partialSum[tid] = 0.0f;

        for (int i = tid; i < nx; i += blockDim.x)
        {
            partialSum[tid] += recon[d * nx + i];
        }

        __syncthreads(); // Ensure all threads have computed their partial sum

        // Reduce partial sums within the block
        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
        {
            if (tid < stride)
            {
                partialSum[tid] += partialSum[tid + stride];
            }
            __syncthreads();
        }

        sum = partialSum[0]; // Only thread 0 has the full sum

        if (tid == 0)
        {
            float upd = data[p * dx + d] - sum;
            for (int i = 0; i < nx; ++i)
            {
                recon[d * nx + i] += upd;
            }
        }
    }
}