#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_sum_dist_compute(int dy, int dx, int nx, int ny, const int32_t* ones, uint32_t* sum_dist, int p)
{
    int nx0      = blockIdx.x * blockDim.x + threadIdx.x;
    int nxstride = blockDim.x * gridDim.x;
    int dx0      = blockIdx.y * blockDim.y + threadIdx.y;
    int dxstride = blockDim.y * gridDim.y;
    int dy0      = blockIdx.z * blockDim.z + threadIdx.z;
    int dystride = blockDim.z * gridDim.z;

    for(int s = dy0; s < dy; s += dystride)
    {
        for(int d = dx0; d < dx; d += dxstride)
        {
            uint32_t* _sum_dist = sum_dist + (s * nx * ny) + (d * nx);
            const int32_t* _ones = ones + (d * nx);
            
            __shared__ int shared_nx[1024];  // Use shared memory to enhance data locality. Adjust size if necessary.
            int tid = threadIdx.x;
            for (int i = tid; i < nx; i += blockDim.x)  // Cooperative loading of data into shared memory.
            {
                shared_nx[i] = (_ones[i] > 0) ? 1 : 0;
            }
            __syncthreads();  // Synchronize to make sure all shared data is loaded.
            
            for(int n = nx0; n < nx; n += nxstride)
            {
                if (shared_nx[n] > 0)
                {
                    atomicAdd(&_sum_dist[n], 1);
                }
            }
        }
    }
}