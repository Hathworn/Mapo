#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_rotate_internal_kernel(float* dst, const float* src, float theta, const int nx, const int ny)
{
    // Precompute trigonometric values for better performance
    float cos_theta = cosf(theta);
    float sin_theta = sinf(theta);

    int src_size = nx * ny;
    float xoff = (0.5f * nx) - 0.5f;
    float yoff = (0.5f * ny) - 0.5f;

    int j0 = blockIdx.x * blockDim.x + threadIdx.x;
    int jstride = blockDim.x * gridDim.x;

    for (int j = j0; j < ny; j += jstride)
    {
        for (int i = 0; i < nx; ++i)
        {
            // Avoid recomputing rx and ry multiple times
            float rx = float(i) - xoff;
            float ry = float(j) - yoff;

            // Use precomputed cos_theta and sin_theta
            float tx = rx * cos_theta - ry * sin_theta;
            float ty = rx * sin_theta + ry * cos_theta;

            float x = (tx + xoff);
            float y = (ty + yoff);

            int rz = j * nx + i;
            auto index = [&](int _x, int _y) { return _y * nx + _x; };

            int x1 = floorf(x);
            int y1 = floorf(y);
            int x2 = x1 + 1;
            int y2 = y1 + 1;

            float fxy1 = 0.0f;
            float fxy2 = 0.0f;

            // Handle boundary conditions with checks
            int ixy11 = index(x1, y1);
            int ixy21 = index(x2, y1);
            int ixy12 = index(x1, y2);
            int ixy22 = index(x2, y2);

            if (ixy11 >= 0 && ixy11 < src_size)
                fxy1 += (x2 - x) * src[ixy11];
            if (ixy21 >= 0 && ixy21 < src_size)
                fxy1 += (x - x1) * src[ixy21];
            if (ixy12 >= 0 && ixy12 < src_size)
                fxy2 += (x2 - x) * src[ixy12];
            if (ixy22 >= 0 && ixy22 < src_size)
                fxy2 += (x - x1) * src[ixy22];

            dst[rz] += (y2 - y) * fxy1 + (y - y1) * fxy2;
        }
    }
}