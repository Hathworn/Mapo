#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Use blockDim.x directly for better readability
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;

    // Initialize each hiprand state with a unique sequence number
    hiprand_init(seed, tidx, 0, &state[tidx]);
}