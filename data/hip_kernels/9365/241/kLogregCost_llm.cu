#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    if (tx < numCases) {
        const int label = __float2int_rd(labels[tx]);  // Use fast rounding conversion
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        if (labelp != maxp) {
            correctProbs[tx] = 0.0f;
        } else {
            int numMax = 0;
            for (int i = 0; i < numOut; i++) {
                numMax += (probs[i * numCases + tx] == maxp);  // Simplified loop condition by using parentheses
            }
            correctProbs[tx] = 1.0f / __int2float_rn(numMax);  // Use precise conversion
        }
    }
}