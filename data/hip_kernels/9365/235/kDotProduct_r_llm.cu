#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];

    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float local_sum = 0.0f; // Use local sum to reduce memory access contention

    if (eidx < numElements) {
        for (; eidx < numElements; eidx += gridDim.x * DP_BLOCKSIZE) {
            local_sum += a[eidx] * b[eidx];
        }
    }
    shmem[threadIdx.x] = local_sum; // Only write once to shared memory
    __syncthreads();

    // Use a loop for reduction to enhance readability and maintainability
    for (uint stride = DP_BLOCKSIZE / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Final sum written to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}