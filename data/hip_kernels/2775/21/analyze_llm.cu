#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void analyze(const float *input, float *sum, int numElements) {
    // Use shared memory for partial sums to reduce contention on global memory
    __shared__ float partialSum[256]; // assuming blockDim.x = 256
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize shared memory
    partialSum[threadIdx.x] = (i < numElements) ? input[i] : 0.0f;

    // Synchronize threads to ensure shared memory is initialized
    __syncthreads();

    // Reduction in shared memory
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            partialSum[threadIdx.x] += partialSum[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Write result for this block to global sum, using atomic add
    if (threadIdx.x == 0) {
        atomicAdd(sum, partialSum[0]);
    }
}