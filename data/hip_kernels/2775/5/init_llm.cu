#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(uint32_t *v, uint32_t val, uint32_t n) {
    // Use shared memory to reduce global memory accesses
    extern __shared__ uint32_t s[];
    
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        // Load val into shared memory
        s[threadIdx.x] = val;
        __syncthreads();
        
        // Write from shared memory to global memory
        v[i] = s[threadIdx.x];
        
        // Move printf outside loop
        if (i == 0) {
            printf("init\n");
        }
    }
}