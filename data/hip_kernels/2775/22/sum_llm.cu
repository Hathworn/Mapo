#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(const float *input, float *output, int numElements) {
    // Use threadIdx.x to process elements in parallel
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    float val = 0.f;
    
    // Ensure index is within bounds
    if (index < numElements) {
        for (int i = index; i < numElements; i += blockDim.x * gridDim.x) {
            val += input[i];
        }
    }
    
    // Use shared memory to reduce sum
    __shared__ float sharedData[256];  // Assumes blockDim.x <= 256
    sharedData[threadIdx.x] = val;
    __syncthreads();
    
    // Reduce using block threads
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            sharedData[threadIdx.x] += sharedData[threadIdx.x + s];
        }
        __syncthreads();
    }
    
    // Write result of block sum to global output
    if (threadIdx.x == 0) {
        atomicAdd(output, sharedData[0]);
    }
}