#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pi_random(float* x, float* y, int* global_count) {
    __shared__ int counts[nthreads];

    int globalId = blockIdx.x * blockDim.x + nitemsperthread * threadIdx.x;

    int thread_count=0;
    for (int i=0; i<nitemsperthread; i++) {
        // Check boundary condition first to minimize wasted operations
        if (globalId+i < nsamples) {
            float x_val = x[globalId+i];
            float y_val = y[globalId+i];
            // Avoid repeated arithmetic operations
            if (x_val * x_val + y_val * y_val < 1.0f) {
                thread_count++;
            }
        }
    }

    counts[threadIdx.x] = thread_count;
    __syncthreads();

    if (threadIdx.x == 0) {
        int block_count = 0;
        // Accumulate results using a single thread after synchronization
        for (int i=0; i<nthreads; i++) {
            block_count += counts[i];
        }
        global_count[blockIdx.x] = block_count;
    }
}