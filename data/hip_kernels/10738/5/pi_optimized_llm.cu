#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pi_optimized(float* x, float* y, int* global_count) {
    __shared__ int counts[nthreads];

    // Calculate global index
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;

    int thread_count = 0;
    for (int i = 0; i < nitemsperthread; i++) {
        int idx = globalId + (i * nthreads * nblocks);
        if (idx < nsamples) {
            // Optimized condition checking for circle inclusion
            if (x[idx] * x[idx] + y[idx] * y[idx] < 1.0f) {
                thread_count++;
            }
        }
    }

    // Store thread count in shared memory
    counts[threadIdx.x] = thread_count;
    __syncthreads();

    // Accumulate counts only once per block
    if (threadIdx.x == 0) {
        int block_count = 0;
        for (int i = 0; i < nthreads; i++) {
            block_count += counts[i];
        }
        // Store block count in global memory
        global_count[blockIdx.x] = block_count;
    }
}