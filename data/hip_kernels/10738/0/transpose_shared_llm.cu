#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

// parameter describing the size of matrix A
const int rows = 4096;
const int cols = 4096;

const int BLOCK_SIZE = 16;

// transpose shared kernel

// transpose kernel

__global__ void transpose_shared(float* a, float* b) {
    __shared__ float result[BLOCK_SIZE][BLOCK_SIZE+1]; // Avoid bank conflict

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int width = gridDim.x * blockDim.x;
    int height = gridDim.y * blockDim.y;

    if (x < height && y < width) {
        result[threadIdx.y][threadIdx.x] = a[y * height + x]; // Swap indices
    }
    __syncthreads();
    if (x < width && y < height) {
        b[y * width + x] = result[threadIdx.x][threadIdx.y]; // Correct indexing
    }
}