#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int* a, int* b, int* c) {
    // Optimization note: use a local variable to reduce global memory accesses.
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Perform addition and store result in a local register variable first.
    int sum = a[id] + b[id];
    
    // Write result back to global memory.
    c[id] = sum;
}