#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int* a, int* b, int* c) {
// calculate global id
int id = blockIdx.x * blockDim.x + threadIdx.x;

// perform calculation
c[id] = a[id] + b[id];
}