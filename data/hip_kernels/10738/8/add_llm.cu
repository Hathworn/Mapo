#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add(int *a, int *b, int *c, int columns, int rows)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Use a single calculation for global index
    int idy = blockIdx.y * blockDim.y + threadIdx.y; // Use a single calculation for global index
    int index = idy * columns + idx; // Calculate index based on 2D grid

    if (idx < columns && idy < rows) { // Ensure within bounds before accessing memory
        c[index] = a[index] + b[index]; // Perform addition
    }
}