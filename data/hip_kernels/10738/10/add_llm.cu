#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add(int *a, int *b, int *c, int columns, int rows)
{
    // Use shared memory to cache input data
    __shared__ int tileA[32][32];
    __shared__ int tileB[32][32];

    // Calculate global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Load data into shared memory if within bounds
    if (x < columns && y < rows)
    {
        int index = y * columns + x;
        tileA[threadIdx.y][threadIdx.x] = a[index];
        tileB[threadIdx.y][threadIdx.x] = b[index];

        // Synchronize to ensure data is loaded
        __syncthreads();

        // Perform computation using shared memory
        c[index] = tileA[threadIdx.y][threadIdx.x] + tileB[threadIdx.y][threadIdx.x];
    }
}