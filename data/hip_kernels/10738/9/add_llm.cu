#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use __restrict__ to tell the compiler that pointers don't alias
__global__ void add(const int* __restrict__ a, const int* __restrict__ b, int* __restrict__ c) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    // Utilize shared memory for coalesced memory access (assumes blockDim.x <= 1024)
    __shared__ int shared_a[1024];
    __shared__ int shared_b[1024];

    // Load elements into shared memory
    shared_a[threadIdx.x] = a[id];
    shared_b[threadIdx.x] = b[id];
    __syncthreads();

    // Perform addition using shared memory
    c[id] = shared_a[threadIdx.x] + shared_b[threadIdx.x];
}