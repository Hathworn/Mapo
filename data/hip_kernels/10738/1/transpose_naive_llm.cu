#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

// parameter describing the size of matrix A
const int rows = 4096;
const int cols = 4096;

const int BLOCK_SIZE = 16;

// transpose shared kernel

// transpose kernel

__global__ void transpose_naive(float* a, float* b) {
    // Calculate global indices
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate accurate width and height
    int width = gridDim.x * blockDim.x;
    int height = gridDim.y * blockDim.y;

    // Boundary check to ensure no out-of-bounds access
    if (x < width && y < height) {
        b[y * width + x] = a[x * height + y];  // Access pattern optimized for coalesced memory
    }
}