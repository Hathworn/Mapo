#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NormalizeOutput(const int num_elements, const int* __restrict__ original, int64_t* __restrict__ to_normalize, int64_t batch_index, int64_t class_index) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop over elements using stride to ensure all elements are processed
    for (; idx < num_elements; idx += blockDim.x * gridDim.x) {
        // Access memory by stride and store normalized values
        to_normalize[idx * 3] = batch_index;
        to_normalize[idx * 3 + 1] = class_index;
        to_normalize[idx * 3 + 2] = static_cast<int64_t>(original[idx]);
    }
}