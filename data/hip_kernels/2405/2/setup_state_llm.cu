#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function by launching multiple threads
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate index for each thread
    hiprand_init(seed, idx, 0, &state[idx]); // Initialize state for each thread
}