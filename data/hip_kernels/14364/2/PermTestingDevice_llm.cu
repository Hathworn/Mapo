#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void PermTestingDevice(int numPermutations, int N, int V, int N_gp1, double *dataDevice, int *permutationsDevice, double *MaxTDevice)
{
    int threadId = threadIdx.x + (blockIdx.x * blockDim.x); // Current Permutation
    int permutationsStart = threadId * N;
    int N_gp2 = N - N_gp1; // Size of group 2
    double MaxT = 0;
    
    extern __shared__ double sharedMem[]; // Shared memory for voxel data
    double *group1Sums = sharedMem;
    double *group2Sums = &sharedMem[blockDim.x];
    double *group1SumSq = &sharedMem[2 * blockDim.x];
    double *group2SumSq = &sharedMem[3 * blockDim.x];

    for (int i = 0; i < V; i++)
    {
        double group1Sum = 0;
        double group2Sum = 0;
        double group1SumSquared = 0;
        double group2SumSquared = 0;

        // Calculate group 1 statistics
        for (int j = 0; j < N_gp1; j++)
        {
            int currSubject = permutationsDevice[permutationsStart + j] - 1;
            double voxelStatistic = dataDevice[currSubject * V + i];
            group1Sum += voxelStatistic;
            group1SumSquared += voxelStatistic * voxelStatistic;
        }

        // Calculate group 2 statistics
        for (int j = N_gp1; j < N; j++)
        {
            int currSubject = permutationsDevice[permutationsStart + j] - 1;
            double voxelStatistic = dataDevice[currSubject * V + i];
            group2Sum += voxelStatistic;
            group2SumSquared += voxelStatistic * voxelStatistic;
        }

        // Compute means and variances
        double group1Mean = group1Sum / N_gp1;
        double group2Mean = group2Sum / N_gp2;
        double group1Var = (group1SumSquared / N_gp1) - (group1Mean * group1Mean);
        double group2Var = (group2SumSquared / N_gp2) - (group2Mean * group2Mean);

        // Compute t-statistics
        double meanDifference = group1Mean - group2Mean;
        double denominator = sqrt((group1Var / N_gp1) + (group2Var / N_gp2));
        double tStat = meanDifference / denominator;

        // Update maximum t-statistic
        if (tStat > MaxT)
        {
            MaxT = tStat;
        }
    }

    MaxTDevice[threadId] = MaxT;
}