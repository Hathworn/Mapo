#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void doGPUWork(int numData, int *data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (idx < numData) { // Check global thread index
        data[idx] = idx; // Use global thread index
    }
}