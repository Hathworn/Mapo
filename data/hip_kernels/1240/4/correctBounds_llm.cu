#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void correctBounds(double *d_ub, double *d_lb, int nRxns, double *d_prevPoint, double alpha, double beta, double *d_centerPoint, double *points, int pointsPerFile, int pointCount, int index) {
    // Calculate global thread index
    int newindex = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Process elements in stride to cover all nRxns
    for (int i = newindex; i < nRxns; i += stride) {
        // Use local variable to reduce global memory access
        double pointValue = points[pointCount + pointsPerFile * i];
        
        // Check bounds and correct point value
        if (pointValue > d_ub[i]) {
            pointValue = d_ub[i];
        } else if (pointValue < d_lb[i]) {
            pointValue = d_lb[i];
        }

        // Update point and store the result
        points[pointCount + pointsPerFile * i] = pointValue;
        d_prevPoint[nRxns * index + i] = pointValue;
        d_centerPoint[nRxns * index + i] = alpha * d_centerPoint[nRxns * index + i] + beta * pointValue;
    }
}