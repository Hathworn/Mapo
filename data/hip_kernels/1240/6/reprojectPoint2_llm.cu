#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reprojectPoint2(double *d_N, int nRxns, int istart, double *d_umat, double *points, int pointsPerFile, int pointCount, int index) {
    // Calculate the global index for the current thread
    int newindex = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Iterate through points using stride for better global memory access pattern
    for (int i = newindex; i < nRxns; i += stride) {
        // Initialize point to zero
        points[pointCount + pointsPerFile * i] = 0;
        // Use loop unrolling to improve performance
        #pragma unroll
        for (int j = 0; j < nRxns - istart; j++) {
            // Accumulate product of d_N and d_umat
            points[pointCount + pointsPerFile * i] += d_N[j * nRxns + i] * d_umat[nRxns * index + j];
        }
    }
}