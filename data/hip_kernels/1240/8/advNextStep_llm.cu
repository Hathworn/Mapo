#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void advNextStep(double *d_prevPoint, double *d_umat, double d_stepDist, int nRxns, double *points, int pointsPerFile, int pointCount, int index) {
    // Calculate global thread index for accessing array elements
    int newindex = blockIdx.x * blockDim.x + threadIdx.x;
    // Loop with stride to cover all elements
    for (int i = newindex; i < nRxns; i += blockDim.x * gridDim.x) {
        // Compute the next step directly and store in points array
        points[pointCount + pointsPerFile * i] = d_prevPoint[nRxns * index + i] + d_stepDist * d_umat[nRxns * index + i];
    }
}