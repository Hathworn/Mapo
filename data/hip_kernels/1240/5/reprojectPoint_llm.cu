#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reprojectPoint(double *d_N, int nRxns, int istart, double *d_umat, double *points, int pointsPerFile, int pointCount, int index){
    int newindex = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    while (newindex < nRxns - istart) {
        // Initialize d_umat element to zero
        double sum = 0.0;

        // Compute t(N) * Pt
        for (int j = 0; j < nRxns; j++) {
            sum += d_N[j + newindex * nRxns] * points[pointCount + pointsPerFile * j];
        }

        // Store the computed sum
        d_umat[nRxns * index + newindex] = sum;

        // Increment by stride for next potential index
        newindex += stride;
    }
}