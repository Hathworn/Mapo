#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function
__global__ void findMaxAbs(int nRxns, double *d_umat2, int nMets, int *d_rowVec, int *d_colVec, double *d_val, int nnz, double *points, int pointsPerFile, int pointCount, int index) {
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use loop unrolling for improved performance
    for (; k < nnz; k += blockDim.x * gridDim.x) {
        double point_val = points[pointCount + pointsPerFile * d_colVec[k]];
        d_umat2[nMets * index + d_rowVec[k]] += d_val[k] * point_val;
    }
}