#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod, const size_t lod_len, size_t* out_lod0) {
    // Utilize shared memory for faster access
    extern __shared__ size_t shared_num_erased[];

    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Load num_erased into shared memory
    if (index < lod_len) {
        shared_num_erased[index] = num_erased[in_lod[index]];
    }

    // Synchronize threads to ensure shared memory is fully populated
    __syncthreads();

    // Calculate output using shared memory for optimization
    if (index < lod_len) {
        out_lod0[index] = in_lod[index] - shared_num_erased[index];
    }
}