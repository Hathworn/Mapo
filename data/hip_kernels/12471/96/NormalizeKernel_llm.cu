#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit if outside image bounds
    if (i >= h || j >= w) return;

    const int pos = i * s + j;
    float scale = normalization_factor[pos];

    // Avoid branch by adding small value to scale
    float invScale = 1.0f / (scale + 1e-7f);

    // Perform normalization
    image[pos] *= invScale;
}