#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate point index
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds
    if (ptidx < npoints)
    {
        // Load loc data into shared memory to optimize access
        __shared__ short2 shared_loc[256]; // Assuming max blockDim.x is 256
        shared_loc[threadIdx.x] = loc_[ptidx];
        __syncthreads();

        short2 loc = shared_loc[threadIdx.x];

        // Perform scaling and write to output
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}