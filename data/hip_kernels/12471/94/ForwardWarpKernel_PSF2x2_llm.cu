#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Compute pixel coordinates and deltas
    float cx = u[flow_row_offset + j] * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + static_cast<float>(i) + 1.0f;
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);
    
    // Convert to integers
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);

    float value = src[image_row_offset + j];
    float weight;

    // Cache bounds to avoid repetitive checks
    bool inBoundsBottomLeft = !(tx < 0 || tx >= w || ty < 0 || ty >= h);
    bool inBoundsBottomRight = !(tx + 1 < 0 || tx + 1 >= w || ty < 0 || ty >= h);
    bool inBoundsUpperLeft = !(tx < 0 || tx >= w || ty - 1 < 0 || ty - 1 >= h);
    bool inBoundsUpperRight = !(tx + 1 < 0 || tx + 1 >= w || ty - 1 < 0 || ty - 1 >= h);

    // Fill pixel containing bottom right corner
    if (inBoundsBottomRight)
    {
        weight = dx * dy;
        _atomicAdd(dst + ty * image_stride + (tx + 1), value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + (tx + 1), weight);
    }

    // Fill pixel containing bottom left corner
    if (inBoundsBottomLeft)
    {
        weight = (1.0f - dx) * dy;
        _atomicAdd(dst + ty * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // Fill pixel containing upper left corner
    if (inBoundsUpperLeft)
    {
        weight = (1.0f - dx) * (1.0f - dy);
        _atomicAdd(dst + (ty - 1) * image_stride + tx, value * weight);
        _atomicAdd(normalization_factor + (ty - 1) * image_stride + tx, weight);
    }

    // Fill pixel containing upper right corner
    if (inBoundsUpperRight)
    {
        weight = dx * (1.0f - dy);
        _atomicAdd(dst + (ty - 1) * image_stride + (tx + 1), value * weight);
        _atomicAdd(normalization_factor + (ty - 1) * image_stride + (tx + 1), weight);
    }
}