#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    int stride = blockDim.x * gridDim.x;            // Calculate stride
    for (; i < size; i += stride) {                 // Use global index and stride
        r[i] = x[i] * y[i] + z[i];
    }
}