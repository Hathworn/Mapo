#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelA(float* r, float* x, float* y, float* z, int size)
{
    // Calculate global index for each thread
    int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread works within bounds
    if (globalIndex < size) {
        r[globalIndex] = x[globalIndex] * y[globalIndex] + z[globalIndex];
    }
}