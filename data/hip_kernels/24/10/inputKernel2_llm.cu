#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inputKernel2(float *x, int n, int N)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = iy * NUM_OF_X_THREADS + ix;

    if (idx < N)
    {
        float value = (idx < n) ? ((float)idx * 2) - ((float)idx * (float)idx) : 0;
        // Use a single loop for initialization
        for(int i = 0; i < N; i++)
        {
            x[idx * N + i] = (i == 0) ? value : 0;
        }
    }
}