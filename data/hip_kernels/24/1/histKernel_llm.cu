#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void histKernel(char *inData, long size, unsigned int *histo)
{
    __shared__ unsigned int temp[BIN_COUNT][BIN_COUNT];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int offset = blockDim.x * gridDim.x;

    // Initialize shared memory for current thread
    for (int i = 0; i < BIN_COUNT; ++i)
        temp[i][threadIdx.x] = 0;

    __syncthreads();

    // Accumulate local histogram in shared memory
    while(tid < size) {
        atomicAdd(&temp[(int)inData[tid]][threadIdx.x], 1);
        tid += offset;
    }

    __syncthreads();

    // Reduce local histograms to global memory
    for (int i = 0; i < BIN_COUNT; ++i)
        atomicAdd(&(histo[i]), temp[i][threadIdx.x]);
}