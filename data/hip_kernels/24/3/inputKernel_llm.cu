#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inputKernel(float *x, int N)
{
    int idx = blockIdx.y * blockDim.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.y * blockDim.x + threadIdx.x; // Optimize index calculation

    if (idx < N)
        x[idx] += (float)idx; // Simplify addition operation
}