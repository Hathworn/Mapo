#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inputKernel(float *x, int n, int N)
{
    // Compute global thread IDs
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = iy * NUM_OF_X_THREADS + ix;
    
    // Use shared memory for temporary storage if needed
    // __shared__ float temp[NUM_OF_X_THREADS * blockDim.y];

    if (idx < N) // Check if within bounds
    {
        // Initialize x[idx * N] or x[idx]
        x[idx*N] = (idx < n) ? (float)idx : 0;

        int i;
        for(i=1; i<N; i++)
        {
            x[idx*N + i] = 0; // Zero-fill other elements in the row
        }
    }
}