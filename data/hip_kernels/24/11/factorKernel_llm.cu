#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void factorKernel(float *w, int N)
{
    int ix  = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = ix * 2;
    int izx = N + idx;

    // Use constant memory for Pi and precompute aw
    const float pi = 3.141592653589793238462643383;
    float aw = (2.0 * pi) / N;
    
    if (idx < N) {
        float arg = aw * ix;  // Compute only when needed within boundary
        float cos_val = cos(arg);
        float sin_val = sin(arg);
        
        // Load values once and reuse
        w[idx] = cos_val;
        w[idx + 1] = sin_val;
        w[izx] = -cos_val;
        w[izx + 1] = -sin_val;
    }
}