#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void twiddleRealKernel(float *wr, float *w, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use loop unrolling to improve performance
    for (; idx < N * N; idx += stride) {
        int row = idx / N;
        int col = idx % N;

        if (col == 0 || row == 0) {
            wr[idx] = 1.0f;
        } else {
            int index = (row * col) % N;
            wr[idx] = w[index * 2];
        }
    }
}