#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void twiddleRealKernel(float *wr, float *w, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for faster access
    extern __shared__ float shared_w[];

    // Load data into shared memory
    int threadIndex = threadIdx.x;
    for (int stride = threadIndex; stride < N; stride += blockDim.x) {
        shared_w[stride] = w[stride * 2];
    }
    __syncthreads();

    if (idx < N) {
        int baseIdx = idx * N;
        wr[baseIdx] = 1;
        for (int i = 1; i < N; i++) {
            int index = (idx * i) % N;
            wr[baseIdx + i] = (idx == 0) ? 1 : shared_w[index];
        }
    }
}