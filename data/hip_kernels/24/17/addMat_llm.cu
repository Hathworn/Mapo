#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addMat(float *a, float *b, float *add, int N)
{
    // Calculate the row and column index of the element
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the row and column are within the matrix boundaries
    if (row < N && col < N)
    {
        // Compute the index and perform addition
        int idx = row * N + col;
        add[idx] = a[idx] + b[idx];
    }
}