#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dotProdKernel(float *a, float *b, float *ab, int N)
{
    // Calculate the index within bounds
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        // Perform parallel dot product operation 
        ab[idx] = a[idx] * b[idx];
    }
}