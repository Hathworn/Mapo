#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subMat(float *a, float *b, float *sub, int N)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Calculate total number of threads
    int numThreads = blockDim.x * gridDim.x;

    // Iterate over the elements assigned to this thread
    for (int i = idx; i < N * N; i += numThreads) {
        sub[i] = a[i] - b[i];
    }
}