#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multKernel(float *a, float *b, float *ab, int width)
{
    int tx = threadIdx.x, ty = threadIdx.y;
    int bx = blockIdx.x, by = blockIdx.y;

    // allocate tiles in __shared__ memory
    __shared__ float s_a[TILE_WIDTH][TILE_WIDTH];
    __shared__ float s_b[TILE_WIDTH][TILE_WIDTH];

    // calculate the row & col index to identify element to work on
    int row = by * blockDim.y + ty;
    int col = bx * blockDim.x + tx;
    float result = 0;

    // loop over the tiles of the input in phases
    for (int p = 0; p < width / TILE_WIDTH; ++p)
    {
        // Load data into shared memory with boundary check
        if (row < width && (p * TILE_WIDTH + tx) < width)
            s_a[ty][tx] = a[row * width + (p * TILE_WIDTH + tx)];
        else
            s_a[ty][tx] = 0.0;

        if ((p * TILE_WIDTH + ty) < width && col < width)
            s_b[ty][tx] = b[(p * TILE_WIDTH + ty) * width + col];
        else
            s_b[ty][tx] = 0.0;

        __syncthreads();

        // dot product between row of s_a and col of s_b
        for (int k = 0; k < TILE_WIDTH; ++k)
            result += s_a[ty][k] * s_b[k][tx];

        // sync threads to ensure all elements are processed
        __syncthreads();
    }

    // store the result with boundary check
    if (row < width && col < width)
        ab[row * width + col] = result;
}