#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void factorKernel(float *w, int N)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = ix * 2;

    // Optimized to avoid repeated calculations
    if (idx < N) {
        float arg = (2.0f * 3.1415f * ix) / N;
        float cos_val = cos(arg);
        float sin_val = sin(arg);

        // Direct assignments with precomputed values
        w[idx] = cos_val;
        w[idx + 1] = sin_val;
        w[N + idx] = -cos_val;
        w[N + idx + 1] = -sin_val;
    }
}