#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void divMat(float *a, int N)
{
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a single comparison to optimize boundary conditions
    if (idx < N)
    {
        // Utilize pointer arithmetic for direct access
        a[idx * N] /= N;
    }
}