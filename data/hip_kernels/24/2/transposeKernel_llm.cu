#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeKernel(float *inData, float *outData)
{
    __shared__ float tile[TILE_DIM][TILE_DIM + 1]; // Prevent bank conflicts

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int y = blockIdx.y * TILE_DIM + threadIdx.y;
    int width = gridDim.x * TILE_DIM;

    // Load data into shared memory - coalesced access
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        if (y + j < width && x < width) // Boundary check
            tile[threadIdx.y + j][threadIdx.x] = inData[(y + j) * width + x];
    }

    __syncthreads();

    // Recalculate x, y for transposition
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    // Store to output array - coalesced access
    for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
    {
        if (y + j < width && x < width) // Boundary check
            outData[(y + j) * width + x] = tile[threadIdx.x][threadIdx.y + j];
    }
}