#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void twiddleImgKernel(float *wi, float *w, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to optimize access
    extern __shared__ float shared_w[];

    if (idx < N) {
        // Preload w array into shared memory
        if (threadIdx.x < N) {
            shared_w[threadIdx.x] = w[threadIdx.x * 2 + 1];
        }
        __syncthreads();

        if (idx == 0) {
            for (int i = 0; i < N; i++)
                wi[idx * N + i] = 0;
        } else {
            wi[idx * N + 0] = 0;
            for (int i = 1; i < N; i++) {
                int index = (idx * i) % N;
                wi[idx * N + i] = shared_w[index];
            }
        }
    }
}