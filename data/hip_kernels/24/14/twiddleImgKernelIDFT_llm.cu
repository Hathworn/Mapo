#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void twiddleImgKernelIDFT(float *wi, float *w, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i, index;

    // Use shared memory to reduce global memory access
    __shared__ float shared_w[];

    if (idx < N) {
        if (idx == 0) {
            for (i = 0; i < N; i++) {
                wi[i] = 0; // Pointer indexing to simplify index calculation
            }
        } else {
            wi[idx * N] = 0;
            for (i = 1; i < N; i++) {
                index = (idx * i) % N;

                // Load w into shared memory for faster access
                if (threadIdx.x == 0) {
                    shared_w[i] = w[index * 2 + 1];
                }
                __syncthreads(); // Ensure all threads have loaded w

                // Use shared memory for computation
                wi[idx * N + i] = shared_w[i];
            }
        }
    }
}