#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void poly_div3(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Process only if idx is within bounds
    if (idx < N) {
        float x = poli[idx];

        // Use temporary variable to store complex polynomial expression
        float polynomial = x * (9 + x * (5 + x * (5 + x)));
        poli[idx] = 5 + x * (7 - polynomial) + 5.0 / x;
    }
}