#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencilReadOnly3(float *src, float *dst, int size, int raio, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += raio+1;
    if (idx >= size)
        return;

    float out = 0.0f;

    // Prefetch stencil weights to shared memory for faster access
    __shared__ float sharedStencilWeight[256];  // Adjust size as needed
    if (threadIdx.x < 2 * raio) {
        sharedStencilWeight[threadIdx.x] = stencilWeight[threadIdx.x];
    }
    __syncthreads();

    // Use fewer registers and restrict pointer usage
    float* __restrict src_ptr = src;
    float* __restrict dst_ptr = dst;
    
    #pragma unroll
    for(int i = -raio; i < raio; i++) {
        out += src_ptr[idx + i] * sharedStencilWeight[i + raio];
    }

    dst_ptr[idx] = out;
}