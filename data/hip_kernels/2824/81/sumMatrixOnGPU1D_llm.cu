#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y; // Support for 2D grid

    if (ix < nx && iy < ny) // Ensure both ix and iy are within bounds
    {
        int idx = iy * nx + ix;
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}