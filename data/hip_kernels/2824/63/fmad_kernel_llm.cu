#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fmad_kernel(double x, double y, double *out)
{
    // Use block-level shared memory to store intermediate calculations
    __shared__ double shared_result;
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Only the first thread in the block performs the FMA operation
    if (tid == 0)
    {
        shared_result = x * x + y;
    }

    // Ensure all threads see the updated value
    __syncthreads();

    // The output is only written once, avoiding multiple writes
    if (tid == 0)
    {
        *out = shared_result;
    }
}