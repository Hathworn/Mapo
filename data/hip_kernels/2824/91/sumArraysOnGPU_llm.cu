#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(double *A, double *B, double *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        // Pre-compute repeated calculations and simplify expressions.
        double Ai = A[i];
        double Bi = B[i];

        double Ai_term = 8 * Ai + Ai / 0.2; // Combine terms involving Ai
        double Bi_term = Bi + 5 * Bi * Bi - 9 * Bi * Bi * Bi / 0.4 + 4 * Bi / 123.1; // Combine terms involving Bi
        double Ai_Bi_product = Bi * Ai; // Precompute Bi * Ai

        C[i] = Ai_term + Bi_term - Ai_Bi_product;
    }
}