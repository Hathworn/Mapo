#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poli3(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure index is within bounds and optimize the polynomial calculation
    if (idx < N) {
        float x = poli[idx];
        float x2 = x * x;
        float x4 = x2 * x2;

        // Use pre-computed values to optimize polynomial calculation
        poli[idx] = 5 * (1 + x * (1 + x * (1 + x2)) + x4);
    }
}