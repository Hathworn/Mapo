#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencilConst1(float *src, float *dst, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 11;  // Combine offset calculation into one step
    if (idx < size)  // Reverse the logic for branch optimization
    {
        float out = 0;
        #pragma unroll
        for (int i = -10; i < 10; i++)
        {
            out += src[idx + i] * const_stencilWeight[i + 10];  // Load and compute simultaneously
        }
        dst[idx] = out;  // Assign the computed value
    }
}