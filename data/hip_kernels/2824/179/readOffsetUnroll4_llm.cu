#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void readOffsetUnroll4(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global thread index
    unsigned int i = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    unsigned int k = i + offset;

    // Ensure the loop continues if within bounds
    float a0, a1, a2, a3, b0, b1, b2, b3;
    if (k + 3 * blockDim.x < n)
    {
        // Load data into registers to minimize global memory access latency
        a0 = A[k];
        b0 = B[k];
        a1 = A[k + blockDim.x];
        b1 = B[k + blockDim.x];
        a2 = A[k + 2 * blockDim.x];
        b2 = B[k + 2 * blockDim.x];
        a3 = A[k + 3 * blockDim.x];
        b3 = B[k + 3 * blockDim.x];

        // Perform computations and store results back to C array
        C[i] = a0 + b0;
        C[i + blockDim.x] = a1 + b1;
        C[i + 2 * blockDim.x] = a2 + b2;
        C[i + 3 * blockDim.x] = a3 + b3;
    }
}