#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalCol(float *out, float *in, const int nx, const int ny)
{
    // Optimize block and thread index calculation for better performance
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Boundary check to ensure indices are within matrix limits
    if (ix < nx && iy < ny)
    {
        // Transpose operation
        out[iy * nx + ix] = in[ix * ny + iy];
    }
}