#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global index with offset
    unsigned int k = blockIdx.x * blockDim.x + threadIdx.x + offset;

    // Use a shared memory buffer for better caching (optional, if applicable)
    __shared__ float sA;
    __shared__ float sB;

    // Check bounds before accessing global memory
    if (k < n) {
        sA = A[k];
        sB = B[k];
        __syncthreads();  // Ensure shared memory loading is complete

        C[k - offset] = sA + sB;  // Write result at the correct index
    }
}