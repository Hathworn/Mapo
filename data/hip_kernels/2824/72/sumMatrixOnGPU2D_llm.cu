#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate global index
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Use shared memory for more efficiency
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    // Check bounds and load data into shared memory
    if (ix < nx && iy < ny) {
        unsigned int idx = iy * nx + ix;
        tileA[threadIdx.y][threadIdx.x] = MatA[idx];
        tileB[threadIdx.y][threadIdx.x] = MatB[idx];
    }
    __syncthreads();  // Synchronize threads to ensure all data is loaded

    // Compute and write back result
    if (ix < nx && iy < ny) {
        unsigned int idx = iy * nx + ix;
        MatC[idx] = tileA[threadIdx.y][threadIdx.x] + tileB[threadIdx.y][threadIdx.x];
    }
}