#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop to utilize memory coalescing and increase throughput
    int stride = blockDim.x * gridDim.x;
    for (int idx = i; idx < N; idx += stride) {
        C[idx] = A[idx] + B[idx];
    }
}