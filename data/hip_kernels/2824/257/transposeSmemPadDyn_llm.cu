#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeSmemPadDyn(float *out, float *in, int nx, int ny)
{
    // Static shared memory with padding
    extern __shared__ float tile[];

    // Coordinate in original matrix
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Linear global memory index for original matrix
    unsigned int ti = iy * nx + ix;

    // Thread index in transposed block
    unsigned int row_idx = threadIdx.y * (blockDim.x + IPAD) + threadIdx.x;
    unsigned int col_idx = threadIdx.x * (blockDim.y + IPAD) + threadIdx.y;

    // Coordinate in transposed matrix
    unsigned int trx = blockDim.y * blockIdx.y + threadIdx.x;
    unsigned int tryy = blockDim.x * blockIdx.x + threadIdx.y;

    // Linear global memory index for transposed matrix
    unsigned int to = tryy * ny + trx;

    // Transpose with boundary test
    if (ix < nx && iy < ny)
    {
        // Load data from global memory to shared memory
        tile[row_idx] = in[ti];

        // Thread synchronization
        __syncthreads();

        // Store data to global memory from shared memory
        if (trx < ny && tryy < nx)
        {
            out[to] = tile[col_idx];
        }
    }
}