#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// kernel function optimized by using a conditional check to print only from one thread
__global__ void helloFromGPU()
{
    if (threadIdx.x == 0 && blockIdx.x == 0) // Print once from the first thread in the first block
    {
        printf("Hello World from GPU!\n");
    }
}