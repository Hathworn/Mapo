#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum4K(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Preload the loop condition check to minimize redundant operations
    if (i >= N) return;
    
    float x[4];

    // Using shared memory to improve memory access times
    __shared__ float sharedA[1024], sharedB[1024];
    
    int localIdx = threadIdx.x;
    for (int j = 0; j < 4; j++)
    {
        if (i < N) 
        {
            sharedA[localIdx] = A[i];
            sharedB[localIdx] = B[i];
            
            x[j] = sharedA[localIdx] * sharedA[localIdx];
            C[i] += sharedA[localIdx] * 3 + 20 * sharedB[localIdx] - sharedA[localIdx] * x[j] + x[j] * sharedB[localIdx] * 7;
            
            i += blockDim.x * gridDim.x;
        }
    }
}