#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    // Use shared memory for performance improvement
    __shared__ float sharedA[256];
    __shared__ float sharedB[256];
    
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Load elements into shared memory
    if (i < N) {
        sharedA[threadIdx.x] = A[i];
        sharedB[threadIdx.x] = B[i];
        __syncthreads(); // Synchronize threads within a block

        // Perform computation using shared memory
        C[i] = sharedA[threadIdx.x] + sharedB[threadIdx.x];
    }
}