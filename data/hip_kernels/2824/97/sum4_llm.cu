#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum4(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize the loop by reducing extraneous increment operations
    #pragma unroll
    for (int j = 0; j < 4; j++)
    {
        int idx = i + j * blockDim.x * gridDim.x;
        if (idx < N) {
            C[idx] = A[idx] + B[idx];
        }
    }
}