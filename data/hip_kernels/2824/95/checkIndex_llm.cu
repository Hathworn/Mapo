#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void checkIndex(void)
{
    // Cache thread and block indices to avoid redundant calls
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int tz = threadIdx.z;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int bz = blockIdx.z;
    int bdx = blockDim.x;
    int bdy = blockDim.y;
    int bdz = blockDim.z;
    int gdx = gridDim.x;
    int gdy = gridDim.y;
    int gdz = gridDim.z;

    // Print cached values
    printf("threadIdx:(%d, %d, %d)\n", tx, ty, tz);
    printf("blockIdx:(%d, %d, %d)\n", bx, by, bz);
    printf("blockDim:(%d, %d, %d)\n", bdx, bdy, bdz);
    printf("gridDim:(%d, %d, %d)\n", gdx, gdy, gdz);
}