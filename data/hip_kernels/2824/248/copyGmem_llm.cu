#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyGmem(float *out, float *in, const int nx, const int ny)
{
    // Use shared memory for coalesced access and reduce global memory transactions
    __shared__ float tile[32][32 + 1]; // Avoid bank conflicts by adding padding

    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned int index_in = iy * nx + ix;

    // Load data into shared memory
    if (ix < nx && iy < ny) {
        tile[threadIdx.y][threadIdx.x] = in[index_in];
    }
    __syncthreads();

    // Calculate transposed index
    unsigned int transposed_ix = blockIdx.y * blockDim.y + threadIdx.x;
    unsigned int transposed_iy = blockIdx.x * blockDim.x + threadIdx.y;

    // Write data from shared memory to global memory with transposition
    unsigned int index_out = transposed_iy * nx + transposed_ix;
    if (transposed_ix < ny && transposed_iy < nx) {
        out[index_out] = tile[threadIdx.x][threadIdx.y];
    }
}