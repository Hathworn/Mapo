#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    // Cache thread index and calculate global index once
    unsigned int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = globalIndex + offset;

    // Ensure index calculation to prevent out-of-bound access
    if (k < n) 
    {
        // Perform computation if within bounds
        C[globalIndex] = A[k] + B[k];
    }
}