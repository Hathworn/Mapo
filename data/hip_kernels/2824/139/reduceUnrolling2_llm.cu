#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling2 (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 2;

    // Load data into shared memory for faster access
    extern __shared__ int sdata[];
    if (idx < n) {
        sdata[tid] = g_idata[idx];
        if (idx + blockDim.x < n) {
            sdata[tid] += g_idata[idx + blockDim.x];
        }
    } else {
        sdata[tid] = 0;
    }

    __syncthreads();

    // In-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sdata[tid] += sdata[tid + stride];
        }
        // Synchronize within threadblock
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}