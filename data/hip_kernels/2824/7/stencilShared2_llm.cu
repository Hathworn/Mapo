#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencilShared2(float *src, float *dst, int size, int raio)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float buffer[1024 + 21];

    // Load data into shared memory with bounds checking
    for (int i = threadIdx.x; i < 1024 + 21; i += blockDim.x)
    {
        int globalIdx = idx + i - raio;
        // Ensure global index is within bounds
        if (globalIdx >= 0 && globalIdx < size)
        {
            buffer[i] = src[globalIdx];
        }
        else
        {
            buffer[i] = 0.0f; // Handle out-of-bound reads
        }
    }

    __syncthreads();

    if (idx < size)
    {
        float out = 0.0f;
        #pragma unroll
        for (int i = -raio; i < raio; i++)
        {
            out += buffer[threadIdx.x + raio + i] * const_stencilWeight[i + raio];
        }

        dst[idx] = out;
    }
}