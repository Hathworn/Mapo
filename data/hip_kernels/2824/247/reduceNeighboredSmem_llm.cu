#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define DIM 256 // Assuming DIM is defined

__global__ void reduceNeighboredSmem(int *g_idata, int *g_odata, unsigned int n)
{
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check
    if (idx >= n) return;

    // load data into shared memory
    smem[tid] = g_idata[idx];
    __syncthreads();

    // in-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) 
    {
        if (tid < stride) 
        {
            smem[tid] += smem[tid + stride];
        }
        __syncthreads(); // synchronize within threadblock
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}