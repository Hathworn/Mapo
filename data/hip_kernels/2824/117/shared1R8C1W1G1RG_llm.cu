#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shared1R8C1W1G1RG(float *A, float *B, float *C, const int N)
{
    __shared__ float Smem[512];
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid branch divergence by checking condition globally
    if (i < N) {
        // Use efficient modulo operation by ensuring shared memory access is within bounds
        int index = threadIdx.x + 1;
        index %= 512;
        
        Smem[index] = A[i];
        
        int readIndex = threadIdx.x * 8;
        readIndex %= 512;

        C[i] = Smem[readIndex];
    }
}