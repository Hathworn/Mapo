#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poly_div1(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        // Load from global memory once
        float x = poli[idx];
        
        // Precompute common sub-expressions
        float x2 = x * x;
        float x3 = x2 * x;
        float x4 = x3 * x;
        float x5 = x4 * x;
        
        // Use precomputed expressions and optimize polynomial evaluation
        poli[idx] = 5 + x * (7 - x * (9 + x * (5 + x * (5 + x)))) + x / 5.0;
    }
}