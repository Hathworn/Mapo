#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kmeans4(short int *input, short int *centroids, int *newcentroids, int *counter, const int n)
{
    const int Dim = 4;
    const int i = blockIdx.x * blockDim.x + threadIdx.x * Dim;
    
    if (i < n) {
        // Load point dimensions
        int point[Dim];
        for (int d = 0; d < Dim; ++d) {
            point[d] = input[i + d];
        }

        // Compute squared distances to centroids
        int squaredDistances[4] = {0};
        #pragma unroll
        for (int k = 0; k < 4; ++k) {
            int offset = k * Dim;
            for (int d = 0; d < Dim; ++d) {
                int diff = point[d] - centroids[offset + d];
                squaredDistances[k] += diff * diff;
            }
        }

        // Reduce to find the minimum distance centroid
        int minIdx = 0;
        int minDist = squaredDistances[0];
        for (int k = 1; k < 4; ++k) {
            if (squaredDistances[k] < minDist) {
                minDist = squaredDistances[k];
                minIdx = k;
            }
        }

        // Update new centroids and counters
        int newCentroidIdx = Dim * minIdx;
        for (int d = 0; d < Dim; ++d) {
            atomicAdd(&newcentroids[newCentroidIdx + d], point[d]);
        }
        atomicAdd(&counter[minIdx], 1);
    }
}