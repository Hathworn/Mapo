#include "hip/hip_runtime.h"
```cuda
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poli4(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure idx is within array bounds
    if (idx < N) {
        float x = poli[idx];
        float sqrt_x = sqrt(x);  // Compute sqrt(x) once
        float x2 = x * x;        // Compute x^2 once
        float x3 = x2 * x;       // Compute x^3 once
        
        // Simplified expression with precomputed power terms
        poli[idx] = 5 + 5 * x + 5 * x * sqrt_x + 5 * sqrt_x * x2 + 5 * x * sqrt_x * x2 + 5 * x * sqrt_x * sqrt_x * x2;
    }
}