#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPUshared(float *A, float *B, float *C, const int N)
{
    __shared__ float smem[512];
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize shared memory to zero
    if (threadIdx.x < 512) {
        smem[threadIdx.x] = 0.0f;
    }
    __syncthreads();

    if (i < N) {
        // Accumulate value into shared memory
        atomicAdd(&smem[threadIdx.x], i);
        __syncthreads();

        // Compute result using shared memory
        C[i] = A[i] + B[i] + smem[threadIdx.x];
    }
}