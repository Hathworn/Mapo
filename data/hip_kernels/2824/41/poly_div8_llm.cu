#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poly_div8(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        float x = poli[idx];
        float x_squared = x * x; // Cache x^2
        float temp = 5 + x * (5 + x); // Compute part of the polynomial
        float result = 5 + x * (7 - x * (9 + x * temp)) + 5.0f / x; // Simplify calculation
        poli[idx] = result;
    }
}