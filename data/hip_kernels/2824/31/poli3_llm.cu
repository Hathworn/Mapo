#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poli3(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < N) {
        float x = poli[idx];
        float x2 = x * x; // precompute x^2
        float x4 = x2 * x2; // precompute x^4
        poli[idx] = 5 + 5 * x + 5 * x2 + 5 * x * x2 + 5 * x4 + 5 * x * x4; // optimize polynomial evaluation
    }
}