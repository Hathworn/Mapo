#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poly_div7(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds before processing
    if (idx < N) {
        float x = poli[idx];
        
        // Precompute repeated expressions for optimization
        float x2 = x * x;
        float x3 = x2 * x;
        float x4 = x3 * x;
        
        // Simplified polynomial calculation using precomputed powers
        poli[idx] = 5 + x * (7 - x * (9 + x * (5 + x * (5 + x)))) + 1.0f / x;
    }
}