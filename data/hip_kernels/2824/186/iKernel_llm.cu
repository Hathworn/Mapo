#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iKernel(float *A, float *B, float *C, const int N)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory to reduce global memory accesses
    __shared__ float sA[256];
    __shared__ float sB[256];

    if (i < N)
    {
        // Load data into shared memory
        sA[threadIdx.x] = A[i];
        sB[threadIdx.x] = B[i];
        __syncthreads();

        // Perform computation using shared memory
        C[i] = sA[threadIdx.x] + sB[threadIdx.x];
    }
}