#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumArraysZeroCopy(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for better memory throughput
    __shared__ float shared_A[256];
    __shared__ float shared_B[256];

    if (i < N) {
        // Load data into shared memory
        shared_A[threadIdx.x] = A[i];
        shared_B[threadIdx.x] = B[i];

        __syncthreads();

        // Compute using shared memory
        C[i] = shared_A[threadIdx.x] + shared_B[threadIdx.x];
    }
}