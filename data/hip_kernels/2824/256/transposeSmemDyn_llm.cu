#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeSmemDyn(float *out, float *in, int nx, int ny)
{
    // Dynamic shared memory allocation
    extern __shared__ float tile[];

    // Global coordinates in original matrix
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Linear global memory index for original matrix
    unsigned int ti = iy * nx + ix;

    // Share memory indices within the block
    unsigned int sharedX = threadIdx.x;
    unsigned int sharedY = threadIdx.y;

    // Transposed block coordinates
    unsigned int transIx = blockDim.y * blockIdx.y + sharedX;
    unsigned int transIy = blockDim.x * blockIdx.x + sharedY;

    // Linear global memory index for transposed matrix
    unsigned int to = transIy * ny + transIx;

    // Transpose operation with boundary check
    if (ix < nx && iy < ny)
    {
        // Load data into shared memory from global memory
        tile[sharedY * blockDim.x + sharedX] = in[ti];

        // Synchronize threads for shared memory usage
        __syncthreads();

        // Store transposed data back to global memory
        out[to] = tile[sharedX * blockDim.y + sharedY];
    }
}