#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum10ops(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        float a = A[i];  // Load A[i] once
        float b = B[i];  // Load B[i] once
        // Combine arithmetic operations to reduce instruction count
        C[i] = a + (b * 10 - 4 * a) * b - a * a - 8;
    }
}