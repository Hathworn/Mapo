#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemUnrollDyn(int *g_idata, int *g_odata, unsigned int n)
{
    extern __shared__ int smem[];

    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // Unrolling 4
    int tmpSum = 0;
    if (idx + 3 * blockDim.x < n)
    {
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        tmpSum = a1 + a2 + a3 + a4;
    }
    smem[tid] = tmpSum;
    __syncthreads();

    // In-place reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) 
    {
        if (tid < s)
        {
            smem[tid] += smem[tid + s];
        }
        __syncthreads();
    }

    // Unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}