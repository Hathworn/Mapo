#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shared4R20ops(float *A, float *B, float *C, const int N)
{
    __shared__ float Smem[512];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < N) {
        // Load data into shared memory
        Smem[threadIdx.x] = A[idx];
    }
    __syncthreads();

    if (idx < N) {
        // Use registers for computations
        float Ai = A[idx];
        float Bi = B[idx];
        float term1 = Ai / 3;
        float term2 = 17 * Bi;
        float term3 = -Ai * Ai;
        float term4 = 3 * Bi;
        float term5 = -4 * Ai * Bi;
        float term6 = 7 * Bi * Bi;
        
        // Compute the result
        float x = term1 + term2 + term3 + term4 + term5 + term6;
        
        // Final computation including shared memory accesses
        int s1 = (threadIdx.x + 1) & 511;
        int s2 = (threadIdx.x + 2) & 511;
        int s3 = (threadIdx.x + 3) & 511;
        int s4 = (threadIdx.x + 4) & 511;
        
        C[idx] = x - 8 
               + Smem[s1] * Ai 
               + 4 * Smem[s2] 
               + 3 * Bi * Smem[s3] 
               + Ai * Smem[s4];
    }
}