#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shared4RNops(float *A, float *B, float *C, const int N)
{
    __shared__ float Smem[512];

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        // Load A directly into shared memory for faster access
        Smem[threadIdx.x] = A[i];
        __syncthreads();  // Ensure all data is loaded into shared memory

        // Combine multiple operations to minimize accesses and improve performance
        float a = A[i];
        float b = B[i];
        float x = a * (7 * a - a / 7.0 + b * 6) + b * 19;

        x += a / 3 + 3 * (b - a * x) + x * b * 7;
        
        // Single access point to memory and index calculation optimized
        C[i] = x - 8 + Smem[(threadIdx.x + 1) & 511] * a 
                     + 4 * Smem[(threadIdx.x + 2) & 511] 
                     + 3 * b * Smem[(threadIdx.x + 3) & 511] 
                     + a * Smem[(threadIdx.x + 4) & 511];
    }
}