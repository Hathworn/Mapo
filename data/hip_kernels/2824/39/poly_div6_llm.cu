#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poly_div6(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Simplify polynomials using Horner's method
    if (idx < N) {
        float x = poli[idx];
        float y = __fdividef(1.0f, x);  // Use fast division
        float poly_value = 5.0f + x * (7.0f - x * (9.0f + x * (5.0f + x * (5.0f + x))));
        poli[idx] = poly_value + y;
    }
}