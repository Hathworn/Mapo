#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeOffsetUnroll2(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x; // Adjust calculation for loop unrolling
    unsigned int k = i + offset;

    if (k + blockDim.x < n)
    {
        float a0 = A[i];
        float b0 = B[i];
        float a1 = A[i + blockDim.x];
        float b1 = B[i + blockDim.x];
        
        C[k]            = a0 + b0;  // Use registers to reduce memory access
        C[k + blockDim.x] = a1 + b1;
    }
}