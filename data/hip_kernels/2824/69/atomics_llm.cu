#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void atomics(int *shared_var, int *values_read, int N, int iters)
{
    // Use register to minimize global memory access
    int i, local_var;
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N) return;

    // Read shared_var once, avoid serializing atomicAdd operation
    local_var = atomicAdd(shared_var, 1);
    values_read[tid] = local_var;

    // Loop unrolling to reduce loop overhead
    for (i = 0; i < iters; i += 4)
    {
        if (i + 0 < iters) atomicAdd(shared_var, 1);
        if (i + 1 < iters) atomicAdd(shared_var, 1);
        if (i + 2 < iters) atomicAdd(shared_var, 1);
        if (i + 3 < iters) atomicAdd(shared_var, 1);
    }
}