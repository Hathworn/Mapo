#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU()
{
    // Use thread 0 in block 0 to print to avoid redundant outputs
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        printf("Hello World from GPU!\n");
    }
}