#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poli2(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float x = poli[idx];
        // Use math library to optimize polynomial computation
        float x2 = x * x;
        poli[idx] = 4 * x2 * x + 3 * x2 - 7 * x + 5;
    }
}