#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffsetUnroll2(float *A, float *B, float *C, const int n, int offset)
{
    // Calculating initial index based on block and thread indices
    unsigned int i = blockIdx.x * blockDim.x * 2 + threadIdx.x; 
    unsigned int k = i + offset;

    // Optimized condition check for unrolling
    if (k < n)
    {
        C[i] = A[k] + B[k];
        if (k + blockDim.x < n)
        {
            C[i + blockDim.x] = A[k + blockDim.x] + B[k + blockDim.x];
        }
    }
}