#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeSmem(float *out, float *in, int nx, int ny)
{
    // static shared memory
    __shared__ float tile[BDIMY][BDIMX+1]; // Avoid bank conflicts by padding

    // coordinate in original matrix
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // linear global memory index for original matrix
    unsigned int ti = iy * nx + ix;

    // boundary check
    if (ix < nx && iy < ny) {
        // load data from global memory to shared memory
        tile[threadIdx.y][threadIdx.x] = in[ti];
    }

    // thread synchronization
    __syncthreads();

    // calculate transposed coordinates
    ix = blockDim.y * blockIdx.y + threadIdx.x;
    iy = blockDim.x * blockIdx.x + threadIdx.y;

    // linear global memory index for transposed matrix
    unsigned int to = iy * ny + ix;

    // boundary check
    if (ix < ny && iy < nx) {
        // store data to global memory from shared memory
        out[to] = tile[threadIdx.x][threadIdx.y];
    }
}