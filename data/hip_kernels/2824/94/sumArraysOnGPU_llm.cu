#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // Use shared memory to reduce global memory accesses
    extern __shared__ float sharedMem[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;
    
    if (i < N) {
        // Load data into shared memory
        sharedMem[tid] = A[i];
        __syncthreads();
        
        // Perform computation using shared memory
        C[i] = sharedMem[tid] + B[i];
    }
}