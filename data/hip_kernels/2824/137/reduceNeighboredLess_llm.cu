#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceNeighboredLess (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if(idx >= n) return;

    // in-place reduction in shared memory for better performance
    __shared__ int sdata[1024]; // assuming maximum blockDim.x is 1024
    sdata[tid] = idata[tid];
    __syncthreads();

    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        // convert tid into local array index
        int index = 2 * stride * tid;

        if (index < blockDim.x)
        {
            sdata[index] += sdata[index + stride];
        }

        // synchronize within threadblock
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sdata[0]; // accumulate result from shared memory
}