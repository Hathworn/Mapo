#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArrays(float *A, float *B, float *C, const int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check index within range and perform addition once
    if (idx < N)
    {
        C[idx] = A[idx] + B[idx];
    }
}