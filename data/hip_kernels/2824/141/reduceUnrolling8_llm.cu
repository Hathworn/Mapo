#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceUnrolling8 (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // Unrolling 8
    int sum = 0;
    if (idx + 7 * blockDim.x < n)
    {
        // Use a loop to reduce redundant code
        #pragma unroll
        for (int i = 0; i < 8; i++)
        {
            sum += g_idata[idx + i * blockDim.x];
        }
        g_idata[idx] = sum;  // Store the sum to the first index
    }

    __syncthreads();

    // In-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride) 
        {
            idata[tid] += idata[tid + stride];  // Combine results within a block
        }
        __syncthreads();  // Synchronize within threadblock
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}