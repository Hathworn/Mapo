#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poli1(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        float x = poli[idx];
        // Directly store the result without creating an intermediate variable
        poli[idx] = 3.0f * x * x - 7.0f * x + 5.0f;
    }
}