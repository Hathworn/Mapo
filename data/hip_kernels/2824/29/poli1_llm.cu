#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poli1(float* __restrict__ poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds before accessing the array
    if (idx < N) {
        float x = poli[idx];
        // Calculate the polynomial value and assign
        poli[idx] = fmaf(fmaf(3.0f, x, -7.0f), x, 5.0f);
    }
}