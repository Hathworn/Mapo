#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop unrolling for efficiency
    if (i < N) {
        C[i] = A[i] + B[i];
        if (i + blockDim.x < N) C[i + blockDim.x] = A[i + blockDim.x] + B[i + blockDim.x];
    }
}