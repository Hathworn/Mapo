#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function
__global__ void stencilReadOnly3(float * __restrict__ src, float * __restrict__ dst, int size, const float* __restrict__ stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 11;
    if (idx >= size + 11) return; // Adjust condition to account for initial index offset

    float out = 0;
    #pragma unroll
    for(int i = -10; i < 10; i++)
    {
        out += __ldg(&src[idx+i]) * __ldg(&stencilWeight[i+10]); // Use __ldg for read-only memory access
    }
    dst[idx] = out;
}