#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float d[256];

    if (i < N) {
        // Each thread computes its result directly without using shared memory.
        C[i] = A[i] + B[i];
    }
}