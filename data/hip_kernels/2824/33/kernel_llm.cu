#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float* polynomial, const size_t N) {
    int thread = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary condition
    if (thread >= N) return;

    float x = polynomial[thread];
    // Precompute common values to reduce redundant calculations
    float x2 = x * x; 
    polynomial[thread] = 3 * x2 - 7 * x + 5;
}