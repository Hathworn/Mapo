#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Use shared memory for better memory access efficiency
    extern __shared__ float sharedMemory[];
    float *sharedA = sharedMemory;
    float *sharedB = sharedA + blockDim.x * blockDim.y;

    unsigned int tx = threadIdx.x;
    unsigned int ty = threadIdx.y;
    unsigned int localIdx = ty * blockDim.x + tx;

    if (ix < nx && iy < ny)
    {
        unsigned int idx = iy * nx + ix;

        // Load data into shared memory
        sharedA[localIdx] = MatA[idx];
        sharedB[localIdx] = MatB[idx];

        // Ensure all threads have loaded their data
        __syncthreads();

        // Compute and store the result
        MatC[idx] = sharedA[localIdx] + sharedB[localIdx];
    }
}