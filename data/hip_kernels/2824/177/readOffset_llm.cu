#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void readOffset(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Pre-calculate offset index to avoid redundant computation
    if (i + offset < n) {
        unsigned int k = i + offset;
        C[i] = A[k] + B[k];
    }
}
```
