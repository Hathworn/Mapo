#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shared4R25ops(float *A, float *B, float *C, const int N)
{
    __shared__ float Smem[512];

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        Smem[threadIdx.x] = A[i]; // Load A[i] into shared memory
    }
    __syncthreads();

    if (i < N) {
        // Store commonly repeated B[i] operations in intermediate variables
        float Bi = B[i];
        float Bi2 = Bi * Bi;
        float Ai = A[i];
        float commonTerm = Ai / 3 + 20 * Bi - Ai * Ai;
        
        // Precompute the tan() function only once
        float tanValue = tan(0.2);

        // Combine repetitive arithmetic computations
        float x = tanValue * Bi + commonTerm - 4 * Ai * Bi + 7 * Bi2;
        
        // Simplify access to shared memory using intermediate variables
        float s1 = Smem[(threadIdx.x + 1) % 512];
        float s2 = Smem[(threadIdx.x + 2) % 512];
        float s3 = Smem[(threadIdx.x + 3) % 512];
        float s4 = Smem[(threadIdx.x + 4) % 512];
        
        C[i] = x - 8 + s1 * Ai + 4 * s2 + 3 * Bi * s3 + Ai * s4;
    }
}