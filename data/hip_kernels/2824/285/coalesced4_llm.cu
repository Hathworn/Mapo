#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void coalesced4(float *A, float *C, const int N)
{
    int i = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    if (i < N - 3) {    // Simplified boundary check
        float4 *A4 = reinterpret_cast<float4*>(A); // Use float4 for coalesced access
        float4 *C4 = reinterpret_cast<float4*>(C);
        C4[blockIdx.x * blockDim.x + threadIdx.x] = A4[blockIdx.x * blockDim.x + threadIdx.x];
    }
}