#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencilReadOnly2(float *src, float *dst, int size, int raio, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Adjust index to account for boundary
    idx += raio + 1;
    if (idx >= size)
        return;
    
    // Declare shared memory for stencil weights
    __shared__ float sharedStencilWeight[1024];
    
    // Cache stencil weights to shared memory
    if (threadIdx.x < (2 * raio))
        sharedStencilWeight[threadIdx.x] = stencilWeight[threadIdx.x];

    // Synchronize to ensure all threads have loaded stencil weights
    __syncthreads();

    float out = 0;
    #pragma unroll
    // Use shared memory for stencil weights
    for (int i = -raio; i < raio; i++)
    {
        out += src[idx + i] * sharedStencilWeight[i + raio];
    }
    dst[idx] = out;
}