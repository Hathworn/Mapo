#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function with optimized indexing
__global__ void poli1(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check within valid range and compute polynomial
    if (idx < N) {
        float x = poli[idx];
        poli[idx] = fmaf(fmaf(3.0f, x, -7.0f), x, 5.0f); // Use FMA for improved performance
    }
}