#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void iKernel(float *src, float *dst)
{
    // Use shared memory for reduced global memory access
    extern __shared__ float shared_src[];
    
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int tid = threadIdx.x;

    // Load data into shared memory
    shared_src[tid] = src[idx];
    __syncthreads();

    // Perform computation using shared memory
    dst[idx] = shared_src[tid] * 2.0f;
}