#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shared4R24ops(float *A, float *B, float *C, const int N)
{
    __shared__ float Smem[512];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int localIdx = threadIdx.x;

    // Load data to shared memory
    if (i < N) Smem[localIdx] = A[i];
    __syncthreads();

    if (i < N) {
        float a = A[i];
        float b = B[i];
        
        // Pre-compute reused expressions
        float a_div_3 = a / 3.0f;
        float b_mul_7 = b * 7.0f;
        float common_denom = a / 9.0f + 13.0f * b;
        
        // Arithmetic operations using pre-computed expressions
        float x = (a_div_3 + 17.0f * b - a * a + 3.0f * b - 4.0f * a * b + b * b_mul_7) / common_denom;

        // Result computations using shared memory
        C[i] = x - 8.0f 
             + Smem[(localIdx + 1) % 512] * a 
             + 4.0f * Smem[(localIdx + 2) % 512]
             + 3.0f * b * Smem[(localIdx + 3) % 512] 
             + a * Smem[(localIdx + 4) % 512];
    }
}