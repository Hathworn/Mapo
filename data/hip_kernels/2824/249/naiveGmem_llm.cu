#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void naiveGmem(float *out, float *in, const int nx, const int ny)
{
    // Use shared memory for optimization and eliminate bank conflicts
    __shared__ float tile[TILE_DIM][TILE_DIM+1];

    // matrix coordinate (ix,iy)
    unsigned int ix = blockIdx.x * TILE_DIM + threadIdx.x;
    unsigned int iy = blockIdx.y * TILE_DIM + threadIdx.y;
    
    // linear index
    unsigned int index_in = iy * nx + ix;
    unsigned int index_out = threadIdx.y * TILE_DIM + threadIdx.x;

    // Load data into shared memory
    if (ix < nx && iy < ny)
    {
        tile[threadIdx.y][threadIdx.x] = in[index_in];
    }
    
    // Synchronize threads in the block
    __syncthreads();
    
    // Compute transposed index
    ix = blockIdx.y * TILE_DIM + threadIdx.x;
    iy = blockIdx.x * TILE_DIM + threadIdx.y;
    
    // Write transposed data back to global memory
    if (ix < ny && iy < nx)
    {
        out[iy * ny + ix] = tile[threadIdx.x][threadIdx.y];
    }
}