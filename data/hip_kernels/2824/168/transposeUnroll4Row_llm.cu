#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Row(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    unsigned int ti = iy * nx + ix; // access in rows

    // Early exit for out of bounds threads
    if (ix + 3 * blockDim.x >= nx || iy >= ny) return;

    // Improved memory access by using registers
    float val0 = in[ti];
    float val1 = in[ti + blockDim.x];
    float val2 = in[ti + 2 * blockDim.x];
    float val3 = in[ti + 3 * blockDim.x];

    unsigned int to = ix * ny + iy; // access in columns

    out[to] = val0;
    out[to + ny * blockDim.x] = val1;
    out[to + ny * 2 * blockDim.x] = val2;
    out[to + ny * 3 * blockDim.x] = val3;
}