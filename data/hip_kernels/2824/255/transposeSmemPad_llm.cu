#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeSmemPad(float *out, float *in, int nx, int ny)
{
    // static shared memory with padding
    __shared__ float tile[BDIMY][BDIMX + IPAD];

    // coordinate in original matrix
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // linear global memory index for original matrix
    unsigned int ti = iy * nx + ix;

    // thread index in transposed block
    unsigned int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = bidx / blockDim.y;
    unsigned int icol = bidx % blockDim.y;

    // coordinate in transposed matrix
    unsigned int transposed_ix = blockDim.y * blockIdx.y + icol;
    unsigned int transposed_iy = blockDim.x * blockIdx.x + irow;

    // linear global memory index for transposed matrix
    unsigned int to = transposed_iy * ny + transposed_ix;

    // Transpose with boundary check
    if (ix < nx && iy < ny) {
        // Load data from global memory to shared memory
        tile[threadIdx.y][threadIdx.x] = in[ti];
    }

    // Thread synchronization
    __syncthreads();

    // Transpose and store back to global memory if within bounds
    if (transposed_ix < ny && transposed_iy < nx) {
        out[to] = tile[icol][irow];
    }
}