#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate the global thread ID for x and y dimensions
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    // Check bounds and perform addition
    if (ix < nx && iy < ny) {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}
```
