#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencilReadOnly2(float *src, float *dst, int size, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;
    if (idx >= size)
        return;

    float out = 0;
    // Use shared memory to reduce global memory access
    __shared__ float shSrc[256 + 20]; // example, adjust based on block size

    int localIdx = threadIdx.x + 11;
    // Load shared memory considering boundary checks
    shSrc[localIdx] = src[idx];
    if (threadIdx.x < 10) { // extra loads for halo elements
        shSrc[threadIdx.x] = src[idx - 11];
        shSrc[localIdx + blockDim.x] = src[idx + blockDim.x];
    }
    __syncthreads();

    #pragma unroll
    for (int i = -10; i < 10; i++) {
        out += shSrc[localIdx + i] * stencilWeight[i + 10];
    }

    dst[idx] = out;
}