#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void poli4(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid global memory access for array out of bounds
    if (idx < N) {
        float x = poli[idx];

        // Compute repeated terms once to reduce arithmetic operations
        float sqrt_x = sqrt(x);
        float x_sqrt_x = x * sqrt_x;
        float x_x_sqrt_x = x * x_sqrt_x;
        
        // Combine like terms to reduce the number of operations
        poli[idx] = 5 + 5 * x + 5 * sqrt_x * x + 10 * x_sqrt_x * x + 5 * sqrt_x * x_x_sqrt_x;
    }
}