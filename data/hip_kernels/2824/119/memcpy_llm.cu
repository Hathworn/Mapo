#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void memcpy( float *dst, float *src )
{
    // Calculate initial global index for this thread
    int index = threadIdx.x + 4 * blockIdx.x * blockDim.x;
    
    // Use register allocation to load data from global memory efficiently
    float a0 = src[index];
    float a1 = src[index + blockDim.x];
    float a2 = src[index + 2 * blockDim.x];
    float a3 = src[index + 3 * blockDim.x];

    // Store data back to global memory
    dst[index] = a0;
    dst[index + blockDim.x] = a1;
    dst[index + 2 * blockDim.x] = a2;
    dst[index + 3 * blockDim.x] = a3;
}