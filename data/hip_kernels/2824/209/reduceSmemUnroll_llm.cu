#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // Static shared memory
    __shared__ int smem[DIM];
    
    // Set thread ID
    unsigned int tid = threadIdx.x;
    
    // Global index
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;
    
    // Unrolling 4 blocks
    int localSum = 0;
    
    // Use loop to reduce code size and improve clarity
    #pragma unroll
    for (int i = 0; i < 4; i++) {
        if (idx + i * blockDim.x < n) {
            localSum += g_idata[idx + i * blockDim.x];
        }
    }

    smem[tid] = localSum;
    __syncthreads();

    // Optimize reduction using dynamic check for block size
    #pragma unroll
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            smem[tid] += smem[tid + s];
        }
        __syncthreads();
    }
    
    // Unrolling warp
    if (tid < 32) {
        volatile int *vsmem = smem;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}