#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void host_api_kernel(float *randomValues, float *out, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a single loop iteration with stride of totalThreads
    for (int i = tid; i < N; i += gridDim.x * blockDim.x)
    {
        float rand = randomValues[i];
        rand = rand * 2;
        out[i] = rand;
    }
}