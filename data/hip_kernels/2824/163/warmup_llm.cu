#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Avoid extra calculations in the kernel by pre-calculating the index
    unsigned int index = iy * nx + ix;

    if (ix < nx && iy < ny)
    {
        out[index] = in[index];
    }
}