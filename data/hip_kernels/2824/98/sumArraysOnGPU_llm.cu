#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use loop to handle multiple elements per thread
    for (int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        C[idx] = A[idx] + B[idx];
    }
}