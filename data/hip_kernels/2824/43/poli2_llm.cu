#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poli2(float* __restrict__ poli, const int N) {
    // Calculate the global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is within bounds and computations are only done when necessary
    if (idx < N) {
        float x = poli[idx];
        float x2 = x * x;       // Precompute x^2 to reduce redundant calculations
        poli[idx] = 4 * x * x2 + 3 * x2 - 7 * x + 5;
    }
}