#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum4Man(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Avoid executing out-of-bounds memory accesses
    if (i >= N) return;

    float x[4], a[4], b[4], c[4];

    #pragma unroll
    for (int j = 0; j < 4; ++j) {
        a[j] = A[i];
        b[j] = B[i];
        x[j] = a[j] / 7.0f;
        c[j] = a[j] / 3 + 20 * b[j]; // 17*b[j] + 3*b[j] simplified to 20*b[j]
        x[j] *= a[j]; // Move repeated expression out of loop

        if (j < 3) {
            i += blockDim.x * gridDim.x;
            if (i >= N) break; // Exit loop if out of bounds
        }
    }

    i = blockIdx.x * blockDim.x + threadIdx.x;
    C[i] += c[0] - x[0];

    i += blockDim.x * gridDim.x;
    if (i < N) C[i] += c[1] - x[1];

    i += blockDim.x * gridDim.x;
    if (i < N) C[i] += c[2] - x[2];

    i += blockDim.x * gridDim.x;
    if (i < N) C[i] += c[3] - x[3];
}