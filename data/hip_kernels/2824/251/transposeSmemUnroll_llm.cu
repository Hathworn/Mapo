#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeSmemUnroll(float *out, float *in, const int nx, const int ny)
{
    // Static 1D shared memory with sufficient space
    __shared__ float tile[BDIMY * BDIMX * 2];

    // Coordinate in original matrix
    unsigned int ix = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Linear global memory index for original matrix
    unsigned int ti = iy * nx + ix;

    // 2D thread index in 2D transposed block
    unsigned int bidx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int irow = bidx / blockDim.y;
    unsigned int icol = bidx % blockDim.y;

    // Coordinate in transposed matrix
    unsigned int ix2 = blockIdx.y * blockDim.y + icol;
    unsigned int iy2 = 2 * blockIdx.x * blockDim.x + irow;

    // Linear global memory index for transposed matrix
    unsigned int to = iy2 * ny + ix2;

    if (ix + blockDim.x < nx && iy < ny)
    {
        // Load two rows from global memory to shared memory
        unsigned int row_idx = 2 * threadIdx.y * blockDim.x + threadIdx.x;
        tile[row_idx] = in[ti];
        tile[row_idx + BDIMX] = in[ti + BDIMX];

        // Thread synchronization: ensuring all data is loaded
        __syncthreads();

        // Store two rows to global memory from two columns of shared memory
        unsigned int col_idx = irow * (blockDim.x * 2) + icol; // Fixing index for accessibility
        out[to] = tile[col_idx];
        out[to + ny * BDIMX] = tile[col_idx + BDIMX];
    }
}
```
