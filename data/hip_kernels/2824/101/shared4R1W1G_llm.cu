#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shared4R1W1G(float *A, float *B, float *C, const int N)
{
    __shared__ float Smem[512];

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds before shared memory access and calculations
    if (i < N) {
        Smem[threadIdx.x] = i; // Store the index in shared memory
        
        // Ensure shared memory accesses stay within bounds using modulo operation
        int idx1 = (threadIdx.x + 1) & 511; // Use bitwise AND instead of modulo for better performance
        int idx2 = (threadIdx.x + 2) & 511;
        int idx3 = (threadIdx.x + 3) & 511;
        int idx4 = (threadIdx.x + 4) & 511;
        
        // Accumulate the result from shared memory
        C[i] = Smem[idx1] + Smem[idx2] + Smem[idx3] + Smem[idx4];
    }
}