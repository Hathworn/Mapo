#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Only perform the calculation if 'i' is within bounds
    if (i < N)
    {
        // Simplified arithmetic operation
        C[i] = A[i] + B[i];
    }
}