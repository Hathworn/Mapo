#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void lots_of_double_compute(double *inputs, int N, size_t niters, double *outputs)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    size_t nthreads = gridDim.x * blockDim.x;

    // Utilize loop unrolling and avoid recomputing values
    for ( ; tid < N; tid += nthreads)
    {
        size_t iter;
        double val = inputs[tid];

        #pragma unroll 4 // Unroll loops to increase instruction level parallelism
        for (iter = 0; iter < niters; iter++)
        {
            // Consolidate operations to reduce arithmetic instruction count
            val = ((val - 91.0) / 3.0 + 203.0) / 1.037;
            val = (val - 102.93) / 0.22 + 109.0;
        }

        outputs[tid] = val;
    }
}