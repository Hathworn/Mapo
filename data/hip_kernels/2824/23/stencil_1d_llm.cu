#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_1d(int *in, int *out, int dim) {

    __shared__ int temp[BLOCK_SIZE + 2*RADIUS];

    int lindex = threadIdx.x + RADIUS;
    int gindex = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = gridDim.x * blockDim.x;

    while (gindex < (dim + blockDim.x)) {

        // Efficiently load inputs into shared memory
        if (gindex < dim)
            temp[lindex] = in[gindex];
        else
            temp[lindex] = 0;

        if (threadIdx.x < RADIUS) {
            int left = gindex - RADIUS;
            int right = gindex + blockDim.x;

            // Load halo on the left
            temp[lindex - RADIUS] = (left >= 0) ? in[left] : 0;

            // Load halo on the right
            temp[lindex + blockDim.x] = (right < dim) ? in[right] : 0;
        }

        __syncthreads();

        // Perform the stencil operation
        int result = 0;
        #pragma unroll
        for (int offset = -RADIUS; offset <= RADIUS; offset++) {
            result += temp[lindex + offset];
        }

        // Store the result
        if (gindex < dim)
            out[gindex] = result;

        gindex += stride;

        __syncthreads();
    }
}