#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poly_div5(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        float x = poli[idx];
        
        // Calculate powers of x to minimize repeated calculations
        float x2 = x * x;
        float x3 = x2 * x;
        float x4 = x3 * x;
        float x5 = x4 * x;
        
        // Compute polynomial using precomputed powers
        float result = 5 + x * (7 - x * (9 + x * (5 + x * (5 + x5))));
        
        // Avoid division inside the polynomial expression to prevent unnecessary complexity
        result += 1.0 / x;

        poli[idx] = result;
    }
}