#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shared1R1W1G(float *A, float *B, float *C, const int N)
{
    __shared__ float Smem[512];

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use conditional to avoid out-of-bounds access and unnecessary writes
    if (i < N) {
        Smem[threadIdx.x] = i;
        __syncthreads();  // Ensure all writes to shared memory have completed

        // Correct modulo to prevent accessing out-of-bound with small arrays
        C[i] = Smem[(threadIdx.x + 1) & (511)]; 
    }
}