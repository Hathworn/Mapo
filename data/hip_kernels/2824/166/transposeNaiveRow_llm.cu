#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void transposeNaiveRow(float *out, float *in, const int nx, const int ny)
{
    // Use shared memory to reduce global memory accesses
    __shared__ float tile[32][32 + 1]; // +1 to avoid bank conflicts

    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;
    
    if (ix < nx && iy < ny)
    {
        tile[threadIdx.y][threadIdx.x] = in[iy * nx + ix];
    }
    
    __syncthreads();
    
    ix = blockDim.y * blockIdx.y + threadIdx.x; // Transpose indices
    iy = blockDim.x * blockIdx.x + threadIdx.y;
    
    if (ix < ny && iy < nx)
    {
        out[iy * ny + ix] = tile[threadIdx.x][threadIdx.y];
    }
}