#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Use shared memory to reduce global memory access
    extern __shared__ float sharedMem[];
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;
    unsigned int idx = iy * nx + ix;

    if (ix < nx && iy < ny) {
        // Load data into shared memory
        sharedMem[threadIdx.y * blockDim.x + threadIdx.x] = MatA[idx] + MatB[idx];
        __syncthreads(); // Ensure all loads are complete
        
        // Write the result back to global memory
        MatC[idx] = sharedMem[threadIdx.y * blockDim.x + threadIdx.x];
    }
}