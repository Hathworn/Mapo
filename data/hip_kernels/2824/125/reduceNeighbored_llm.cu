#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n) 
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if (idx >= n) return;

    // In-place reduction in shared memory
    for (int stride = 1; stride < blockDim.x; stride *= 2) 
    {
        int index = 2 * stride * tid; // Calculate element index for stride
        if (index < blockDim.x) 
        {
            idata[index] += idata[index + stride]; // Sum pairs of elements
        }
        // Synchronize within threadblock
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}