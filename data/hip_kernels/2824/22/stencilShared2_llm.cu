#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencilShared2(float *src, float *dst, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float buffer[1024 + 21];
    
    // Load data into shared memory more efficiently
    for (int i = threadIdx.x; i < 1024 + 21; i += blockDim.x)
    {
        int global_idx = idx + i - threadIdx.x;
        if (global_idx < size)  // Boundary check
        {
            buffer[i] = src[global_idx];
        }
        else
        {
            buffer[i] = 0.0f;  // Handle out-of-bound condition
        }
    }

    __syncthreads();

    idx += 11;
    if (idx >= size)
        return;

    float out = 0.0f;
    #pragma unroll
    for (int i = -10; i < 10; i++)
    {
        out += buffer[threadIdx.x + 10 + i] * const_stencilWeight[i + 10];
    }
    dst[idx] = out;
}