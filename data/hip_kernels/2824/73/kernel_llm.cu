#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *A, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x * 16;

    // Utilize vectorized memory accesses assuming A and C are aligned
    #pragma unroll
    for (int j = 0; j < 16; j++) {
        if (i + j < N) {
            C[i + j] = A[i + j];
        }
    }
}