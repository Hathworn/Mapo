#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // Calculate thread's global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bounds and perform computation if valid
    if (i < N)
    {
        C[i] = A[i] + B[i];
    }
}