#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poli4(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        float x = poli[idx];
        float sqrt_x = sqrt(x);
        float x_squared = x * x;
        // Reduce redundant calculations
        poli[idx] = 5 + 5 * x + 5 * x * sqrt_x + 5 * sqrt_x * x_squared + 
                    5 * x * sqrt_x * x_squared + 5 * x * sqrt_x * sqrt_x * x_squared;
    }
}