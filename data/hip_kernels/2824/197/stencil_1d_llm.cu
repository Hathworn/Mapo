#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil_1d(float *in, float *out)
{
    // Shared memory
    extern __shared__ float smem[];

    // Index to global memory
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Index to shared memory for stencil calculation
    int sidx = threadIdx.x + RADIUS;

    // Read data from global memory into shared memory
    smem[sidx] = in[idx];

    // Read halo part to shared memory with bounds check
    if (threadIdx.x < RADIUS) {
        smem[sidx - RADIUS] = in[max(0, idx - RADIUS)]; // Ensure reading within bounds
        smem[sidx + BDIM] = in[min(idx + BDIM, gridDim.x * blockDim.x - 1)]; // Ensure reading within bounds
    }

    // Synchronize (ensure all the data is available)
    __syncthreads();

    // Apply the stencil
    float tmp = 0.0f;
    #pragma unroll
    for (int i = 1; i <= RADIUS; i++) {
        tmp += coef[i] * (smem[sidx + i] - smem[sidx - i]);
    }

    // Store the result
    out[idx] = tmp;
}