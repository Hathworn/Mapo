#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void simple_reduction(int *shared_var, int *input_values, int N, int iters)
{
    __shared__ int local_mem[256];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;

    for (int iter = 0; iter < iters; iter++)
    {
        // Load input values into shared memory
        if (tid < N)
        {
            local_mem[local_tid] = input_values[tid];
        }
        else
        {
            local_mem[local_tid] = 0; // Ensure no garbage value
        }

        __syncthreads(); // Synchronize to ensure all values are loaded

        // Perform parallel reduction
        for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
        {
            if (local_tid < stride)
            {
                local_mem[local_tid] += local_mem[local_tid + stride];
            }
            __syncthreads(); // Ensure shared data consistency
        }

        // Write result from the block to global memory
        if (local_tid == 0)
        {
            atomicAdd(shared_var, local_mem[0]);
        }
        __syncthreads(); // Ensure completion before the next iteration
    }
}