#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceInterleaved (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;
    
    // boundary check
    if(idx >= n) return;

    // in-place reduction in global memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        // Unroll last warp to optimize performance
        if (blockDim.x >= 64 && tid < 32 && stride == 32)
        {
            idata[tid] += idata[tid + 32];
            idata[tid] += idata[tid + 16];
            idata[tid] += idata[tid + 8];
            idata[tid] += idata[tid + 4];
            idata[tid] += idata[tid + 2];
            idata[tid] += idata[tid + 1];
        }
        else if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }

        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}