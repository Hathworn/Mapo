#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU2D(float *A, float *B, float *C, int NX, int NY)
{
    // Calculate 1D index from 2D grid and block
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure index is within bounds
    if (ix < NX && iy < NY)
    {
        // Perform addition operation
        unsigned int idx = iy * NX + ix; // Compute index inside the if-statement
        C[idx] = A[idx] + B[idx];
    }
}