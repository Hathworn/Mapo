#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;

    // Ensure kernel is only computing for valid indices
    if (ix < nx && iy < ny) 
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}