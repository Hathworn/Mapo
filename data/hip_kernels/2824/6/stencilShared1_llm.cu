#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencilShared1(float *src, float *dst, int size, int raio)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float buffer[1024 + 21]; // Allocate shared memory with correct size

    // Load data into shared memory with necessary offset handling
    int sharedIndex = threadIdx.x;
    while (sharedIndex < 1024 + 21) {
        int sourceIndex = idx + sharedIndex;
        if (sourceIndex < size) {
            buffer[sharedIndex] = src[sourceIndex];
        }
        sharedIndex += blockDim.x;
    }

    __syncthreads(); // Ensure all threads have loaded their data

    idx += raio + 1;
    if (idx < size) {
        float out = 0.0f;

        #pragma unroll
        for (int i = -raio; i <= raio; i++) {
            out += buffer[threadIdx.x + raio + i] * const_stencilWeight[i + raio];
        }

        dst[idx] = out; // Write result to global memory
    }
}