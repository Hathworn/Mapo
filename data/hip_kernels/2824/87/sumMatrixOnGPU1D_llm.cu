#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU1D(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    if (ix >= nx) return; // Early exit if thread is out of bounds

    // Use loop unrolling for improved performance
    for (int iy = 0; iy < ny; iy += 4)
    {
        int idx = iy * nx + ix;
        MatC[idx] = MatA[idx] + MatB[idx];
        
        if (iy + 1 < ny) // Avoid out of bounds in unrolling
        {
            idx = (iy + 1) * nx + ix;
            MatC[idx] = MatA[idx] + MatB[idx];
        }
        if (iy + 2 < ny) 
        {
            idx = (iy + 2) * nx + ix;
            MatC[idx] = MatA[idx] + MatB[idx];
        }
        if (iy + 3 < ny) 
        {
            idx = (iy + 3) * nx + ix;
            MatC[idx] = MatA[idx] + MatB[idx];
        }
    }
}