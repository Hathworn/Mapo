#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refill_randoms(float *dRand, int N, hiprandState *states)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    hiprandState localState = states[tid]; // Load state into a local variable

    for (int i = tid; i < N; i += stride)
    {
        dRand[i] = hiprand_uniform(&localState); // Use local state for uniform distribution
    }
    
    states[tid] = localState; // Save the state back to global memory
}