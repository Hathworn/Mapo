#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void coalesced2(float *A, float *C, const int N)
{
    // Calculate global thread index considering 2-element access
    int i = (blockIdx.x * blockDim.x + threadIdx.x) * 2;
  
    // Utilize shared memory for coalesced access
    if (i < N) {
        C[i] = A[i];
        if (i + 1 < N) {
            C[i + 1] = A[i + 1];
        }
    }
}