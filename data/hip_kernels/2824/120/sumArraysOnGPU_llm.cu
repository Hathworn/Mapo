#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use memory access coalescing
    int stride = gridDim.x * blockDim.x;
    for (int index = i; index < N; index += stride)
    {
        C[index] = A[index] * B[index];
    }
}