#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void writeOffset(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global thread index
    unsigned int k = blockIdx.x * blockDim.x + threadIdx.x + offset;

    // Ensure all threads in a warp have valid memory accesses
    if (k < n) 
    {
        C[k] = A[k - offset] + B[k - offset]; // Use 'k - offset' to prevent recalculation
    }
}