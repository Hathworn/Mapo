#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shared4R15ops(float *A, float *B, float *C, const int N)
{
    __shared__ float Smem[512];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    float x;

    // Load A into shared memory for improved data reuse
    if (i < N) {
        Smem[threadIdx.x] = A[i];
    }
    __syncthreads();

    if (i < N) {
        // Precompute constant expressions
        float tan_val = tan(0.2);
        
        // Simplify repeated calculations and exploit shared memory usage
        x = tan_val * B[i];
        x += A[i] / 3 + 17 * B[i];
        C[i] = x - 8 
             + Smem[(threadIdx.x + 1) % 512] * A[i]
             + 4 * Smem[(threadIdx.x + 2) % 512] 
             + 3 * Smem[(threadIdx.x + 3) % 512] * B[i] 
             + Smem[(threadIdx.x + 4) % 512] * A[i];
    }
}