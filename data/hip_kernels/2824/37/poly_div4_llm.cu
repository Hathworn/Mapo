#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poly_div4(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        float x = poli[idx];
        float inv_x = 5.0f * __frcp_rn(x); // Fast reciprocal approximation
        poli[idx] = 5.0f + x * (7.0f - x * (9.0f + x * (5.0f + x * (5.0f + x)))) + inv_x;
    }
}