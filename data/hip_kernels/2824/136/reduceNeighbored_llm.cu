#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighbored(int *g_idata, int *g_odata, unsigned int n) {
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if (idx >= n) return;

    // In-place reduction in shared memory for faster access
    for (int stride = 1; stride < blockDim.x; stride *= 2) {
        // Ensure access is not out of bounds
        if (tid + stride < blockDim.x) {
            if ((tid % (2 * stride)) == 0) {
                idata[tid] += idata[tid + stride];
            }
        }
        // Synchronize within threadblock
        __syncthreads();
    }

    // Write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}