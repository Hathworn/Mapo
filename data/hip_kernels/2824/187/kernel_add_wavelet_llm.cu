#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_add_wavelet(float *g_u2, float wavelets, const int nx, const int ny, const int ngpus)
{
    // Calculate global index in x direction
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (ix < nx) { // Ensure within bounds
        // Determine position based on number of GPUs
        int ipos = (ngpus == 2 ? ny - 10 : ny / 2 - 10);
        unsigned int idx = ipos * nx + ix;
        
        // Conditional wavelet addition when ix matches
        if (ix == nx / 2) {
            g_u2[idx] += wavelets;
        }
    }
}