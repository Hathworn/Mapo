#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencilShared1(float *src, float *dst, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float buffer[1024+21];
    
    // Load data into shared memory with coalesced access
    for (int i = threadIdx.x; i < 1024 + 21; i += blockDim.x)
    {
        int bufferIndex = blockIdx.x * blockDim.x + i;
        if (bufferIndex < size)
        {
            buffer[i] = src[bufferIndex];
        }
    }

    __syncthreads();
    
    // Early exit if out of bounds
    idx += 10; // Adjust initial index for stencil calculation

    if (idx >= size)
        return;

    float out = 0;
    #pragma unroll
    for (int i = -10; i < 10; i++)
    {
        out += buffer[threadIdx.x + 10 + i] * const_stencilWeight[i + 10];
    }
    
    dst[idx] = out;
}