#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void random(float *A, float *B, float *C, const int N)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load index from B into a local variable
    int index = __float2int_rd(B[i]);

    // Check bounds and perform the copy
    if (index < N) {
        C[index] = A[index];
    }
}