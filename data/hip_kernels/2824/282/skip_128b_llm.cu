#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void skip_128b(float *A, float *C, const int N)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure correct memory alignment with offset
    i += 32 * (threadIdx.x % 32);

    // Efficient memory access avoiding conditional checks
    if (i < N) {
        C[i] = A[i];
    }
}