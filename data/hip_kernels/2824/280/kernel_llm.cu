#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *g_data, float value)
{
    // Precompute block size and grid size for efficiency
    const int gridSize = blockDim.x * gridDim.x;

    // Stride loop allows handling larger arrays efficiently
    for (int idx = blockIdx.x * blockDim.x + threadIdx.x; idx < gridSize; idx += gridSize)
    {
        g_data[idx] += value;
    }
}