#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPUMix(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate 2D thread index
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y; // Change y to threadIdx.y for 2D block
    unsigned int idx = iy * nx + ix;

    // Guard against out-of-bounds threads
    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}