#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void stencilReadOnly1(float *src, float *dst, int size, int raio, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + raio + 1;
    if (idx >= size)
        return;

    float out = 0.0f;
    // Using unrolled loop for pipelining and reduced loop overhead
    #pragma unroll
    for(int i = -raio; i < raio; i++) {
        out += src[idx+i] * stencilWeight[i+raio];
    }
    dst[idx] = out;
}