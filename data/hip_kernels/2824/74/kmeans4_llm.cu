#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kmeans4(short int *input, short int *centroids, int *newcentroids, int *counter, const int n)
{
    int Dim = 4;
    int i = (blockIdx.x * blockDim.x + threadIdx.x) * Dim;
    if (i < n) {
        // Load point dimensions
        int point[4];
        #pragma unroll
        for (int d = 0; d < Dim; ++d) {
            point[d] = input[i + d];
        }

        // Compute squared distances to centroids
        int distances[4] = {0};
        #pragma unroll
        for (int k = 0; k < 4; ++k) {
            #pragma unroll
            for (int d = 0; d < Dim; ++d) {
                int diff = point[d] - centroids[k * Dim + d];
                distances[k] += diff * diff;
            }
        }
        
        // Find the nearest centroid
        int minIndex = 0;
        int minValue = distances[0];
        #pragma unroll
        for (int k = 1; k < 4; ++k) {
            if (distances[k] < minValue) {
                minValue = distances[k];
                minIndex = k;
            }
        }

        // Atomically update new centroids and counter
        #pragma unroll
        for (int d = 0; d < Dim; ++d) {
            atomicAdd(&(newcentroids[minIndex * Dim + d]), point[d]);
        }
        atomicAdd(&(counter[minIndex]), 1);
    }
}