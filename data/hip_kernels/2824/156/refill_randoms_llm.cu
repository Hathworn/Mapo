#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void refill_randoms(float *dRand, int N, hiprandState *states)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int nthreads = gridDim.x * blockDim.x;

    if (tid < N)
    {
        hiprandState localState = states[tid]; // Use local copy of state
        for (int i = tid; i < N; i += nthreads)
        {
            dRand[i] = hiprand_uniform(&localState); // Generate random number
        }
        states[tid] = localState; // Save state back to global memory
    }
}