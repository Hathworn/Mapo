#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unsafe(int *shared_var, int *values_read, int N, int iters)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= N) return;

    int old, local_shared_var = *shared_var;

    // Store initial value read
    values_read[tid] = local_shared_var;

    // Perform updates with local copy to minimize global memory access
    for (int i = 0; i <= iters; i++) // Run for one extra iteration
    {
        local_shared_var++;
    }

    // Write back the updated value to global memory
    *shared_var = local_shared_var;
}