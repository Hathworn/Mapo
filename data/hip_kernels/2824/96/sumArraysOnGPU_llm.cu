#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // Optimize by calculating global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use grid-stride loop for coalesced memory access and handling larger arrays
    while (i < N) {
        C[i] = A[i] + B[i];
        i += blockDim.x * gridDim.x;
    }
}