#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencil(float *src, float *dst, int size, float *stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += 11;

    // Load data into shared memory to reduce global memory accesses
    __shared__ float sharedSrc[1024 + 20]; // Adjust size based on blockDim.x
    int sharedIdx = threadIdx.x + 10;
    
    if (idx < size) {
        sharedSrc[sharedIdx] = src[idx];
        if (threadIdx.x < 10) {
            // Load halo elements
            sharedSrc[threadIdx.x] = src[idx - 11];
            sharedSrc[sharedIdx + blockDim.x] = src[idx + blockDim.x];
        }
    }
    __syncthreads();

    if (idx >= size)
        return;

    // Use shared array to accumulate the results
    float out = 0;
    #pragma unroll
    for(int i = -10; i < 10; i++)
    {
        out += sharedSrc[sharedIdx + i] * stencilWeight[i + 10];
    }
    dst[idx] = out;
}