#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poli_warp(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds first to avoid unnecessary calculations
    if (idx < N) {
        float x = poli[idx];

        // Refactor polynomial computation for efficiency
        float x2 = x * x;
        float x3 = x2 * x;
        float x4 = x3 * x;

        // Combine terms for less operations
        poli[idx] = 5 + x * (7 - x * (9 + x * (5 + x * (5 + x)))) - 1.0f / x + 3.0f / x2 + x / 5.0f;
    }
}