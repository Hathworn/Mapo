#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceNeighbored (int *g_idata, int *g_odata, unsigned int n)
{
    // Set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // Boundary check
    if (idx >= n) return;

    // In-place reduction in shared memory to reduce global memory access
    __shared__ int shared_idata[1024]; // Assume max blockDim.x is 1024
    shared_idata[tid] = idata[tid];
    __syncthreads();

    // Optimal reduction process using shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            shared_idata[tid] += shared_idata[tid + stride];
        }
        __syncthreads();
    }

    // Write result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = shared_idata[0];
}