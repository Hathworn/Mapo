#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum4(float4 *A, float4 *B, float4 *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < N)
    {
        // Perform vector addition directly using float4 structure
        C[i] = make_float4(A[i].x + B[i].x, A[i].y + B[i].y, A[i].z + B[i].z, A[i].w + B[i].w);
    }
}