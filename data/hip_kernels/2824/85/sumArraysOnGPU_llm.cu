#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // Use shared memory to enhance performance
    extern __shared__ float sharedMemory[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Load inputs into shared memory
    if (idx < N) {
        sharedMemory[tid] = A[idx] * B[idx];
    }
    __syncthreads();

    // Perform operations in shared memory
    if (idx < N) {
        C[idx] = sharedMemory[tid];
    }
}