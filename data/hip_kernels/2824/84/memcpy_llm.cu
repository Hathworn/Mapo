#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void memcpy( float *dst, float *src )
{
    // Calculate index based on block and thread
    int index = threadIdx.x + 4 * blockIdx.x * blockDim.x;
    
    // Use register allocation for optimal memory access
    float a[4];
    
    // Loop unrolling for loading and storing operations
    a[0] = src[index];
    a[1] = src[index + blockDim.x];
    a[2] = src[index + 2 * blockDim.x];
    a[3] = src[index + 3 * blockDim.x];
    
    dst[index] = a[0];
    dst[index + blockDim.x] = a[1];
    dst[index + 2 * blockDim.x] = a[2];
    dst[index + 3 * blockDim.x] = a[3];
}