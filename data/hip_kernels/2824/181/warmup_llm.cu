#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void warmup(float *A, float *B, float *C, const int n, int offset)
{
    // Calculate global thread index
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int k = i + offset;

    // Check if k is in range, and perform computation efficiently
    if (k < n) {
        float a_val = A[i];  // Load A[i] to a register
        float b_val = B[i];  // Load B[i] to a register
        C[k] = a_val + b_val;  // Perform addition
    }
}