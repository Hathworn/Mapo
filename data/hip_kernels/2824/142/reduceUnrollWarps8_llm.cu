#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrollWarps8(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 8 + threadIdx.x;
    
    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 8;

    // unrolling 8 with boundary check
    int sum = 0;
    #pragma unroll 8
    for (int i = 0; i < 8 && (idx + i * blockDim.x) < n; ++i) {
        sum += g_idata[idx + i * blockDim.x];
    }
    // store result in the shared data
    idata[tid] = sum;

    __syncthreads();

    // in-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 32; stride >>= 1) {
        if (tid < stride) {
            idata[tid] += idata[tid + stride];
        }
        // synchronize within threadblock
        __syncthreads();
    }

    // unrolling warp, assumed warp synchronous programming
    if (tid < 32) {
        volatile int *vmem = idata;
        vmem[tid] += vmem[tid + 32];
        vmem[tid] += vmem[tid + 16];
        vmem[tid] += vmem[tid + 8];
        vmem[tid] += vmem[tid + 4];
        vmem[tid] += vmem[tid + 2];
        vmem[tid] += vmem[tid + 1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}