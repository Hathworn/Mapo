#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencilReadOnly1(float *src, float *dst, int size, float* stencilWeight)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 11;
    if (idx >= size) return;

    // Use shared memory to reduce global memory access latency
    __shared__ float sharedStencilWeight[20];
    if (threadIdx.x < 20) {
        sharedStencilWeight[threadIdx.x] = stencilWeight[threadIdx.x];
    }
    __syncthreads();

    float out = 0;
    #pragma unroll
    for(int i = -10; i < 10; i++)
    {
        out += src[idx + i] * sharedStencilWeight[i + 10];
    }
    dst[idx] = out;
}