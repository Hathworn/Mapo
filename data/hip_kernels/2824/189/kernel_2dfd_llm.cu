#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_2dfd(float *g_u1, float *g_u2, const int nx, const int iStart, const int iEnd)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stx = threadIdx.x + NPAD;
    unsigned int idx = ix + iStart * nx;

    __shared__ float line[BDIMX + NPAD2];

    const float alpha = 0.12f;
    float yval[9];

    #pragma unroll
    for (int i = 0; i < 8; i++) yval[i] = g_u2[idx + (i - 4) * nx];

    int iskip = NPAD * nx;
    
    for (int iy = iStart; iy < iEnd; iy++)
    {
        // Load next Y value
        yval[8] = g_u2[idx + iskip];

        // Load surrounding X values into shared memory
        if (threadIdx.x < NPAD)
        {
            line[threadIdx.x] = g_u2[idx - NPAD];
            line[stx + BDIMX] = g_u2[idx + BDIMX];
        }
        line[stx] = yval[4];
        __syncthreads();

        // Perform computation only if within bounds
        if ((ix >= NPAD) && (ix < nx - NPAD))
        {
            // Initialize center point computation
            float tmp = coef[0] * line[stx] * 2.0f;

            // Compute contributions from surrounding points in X and Y
            #pragma unroll
            for (int d = 1; d <= 4; d++)
            {
                tmp += coef[d] * (line[stx - d] + line[stx + d]);
                tmp += coef[d] * (yval[4 - d] + yval[4 + d]);
            }

            // Update the output value based on computed value
            g_u1[idx] = yval[4] + yval[4] - g_u1[idx] + alpha * tmp;
        }

        // Slide Y values down for the next iteration
        #pragma unroll
        for (int i = 0; i < 8; i++)
        {
            yval[i] = yval[i + 1];
        }

        // Advance index for global memory access
        idx += nx;
        __syncthreads();
    }
}