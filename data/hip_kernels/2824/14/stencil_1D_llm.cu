#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCKSIZE 256
#define RADIUS 1

__global__ void stencil_1D(int *in, int *out, int dim) {

    __shared__ int temp[BLOCKSIZE + 2 * RADIUS];

    int lindex = threadIdx.x + RADIUS;
    int gindex = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Optimize boundary checks and simplify logic
    while (gindex < dim) {
        temp[lindex] = (gindex < dim) ? in[gindex] : 0;

        if (threadIdx.x < RADIUS) {
            int left = gindex - RADIUS;
            int right = gindex + blockDim.x;

            temp[lindex - RADIUS] = (left >= 0) ? in[left] : 0;
            temp[lindex + blockDim.x] = (right < dim) ? in[right] : 0;
        }

        __syncthreads();

        // Simplified stencil computation
        int result = 0;
        for (int offset = -RADIUS; offset <= RADIUS; offset++) {
            result += temp[lindex + offset];
        }

        if (gindex < dim) out[gindex] = result;

        gindex += stride;
        __syncthreads();
    }
}