#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copyCol(float *out, float *in, const int nx, const int ny)
{
    // Calculate flattened index in 1D array
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x +
                         (blockDim.y * blockIdx.y + threadIdx.y) * nx;

    // Check if within bounds of the input matrix
    if (index < nx * ny)
    {
        // Direct copy of the element
        out[index] = in[index];
    }
}