#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeNaiveCol(float *out, float *in, const int nx, const int ny)
{
    // Use shared memory to reduce global memory accesses
    __shared__ float tile[32][32 + 1]; // Add padding to avoid bank conflicts

    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    if (ix < nx && iy < ny)
    {
        // Load from global memory to shared memory
        tile[threadIdx.y][threadIdx.x] = in[ix * ny + iy];
    }
    __syncthreads();

    // Transpose within shared memory and write back to global memory
    ix = blockDim.y * blockIdx.y + threadIdx.x; // swap x and y for the transposed position
    iy = blockDim.x * blockIdx.x + threadIdx.y;

    if (ix < ny && iy < nx)
    {
        out[iy * ny + ix] = tile[threadIdx.x][threadIdx.y];
    }
}