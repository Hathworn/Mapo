#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceSmemUnroll(int *g_idata, int *g_odata, unsigned int n)
{
    // static shared memory
    __shared__ int smem[DIM];

    // set thread ID
    unsigned int tid = threadIdx.x;

    // global index, 4 blocks of input data processed at a time
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // unrolling 4 blocks
    int tmpSum = 0;

    // boundary check
    if (idx + 3 * blockDim.x < n) {
        // Efficient memory coalescing
        int a1 = g_idata[idx];
        int a2 = g_idata[idx + blockDim.x];
        int a3 = g_idata[idx + 2 * blockDim.x];
        int a4 = g_idata[idx + 3 * blockDim.x];
        tmpSum = a1 + a2 + a3 + a4;
    }
    smem[tid] = tmpSum;
    __syncthreads();

    // in-place reduction in shared memory using loop to handle various block sizes
    for(int offset = blockDim.x / 2; offset > 32; offset >>= 1) {
        if (tid < offset) smem[tid] += smem[tid + offset];
        __syncthreads();
    }

    // unrolling warp
    if (tid < 32) {
        volatile int* vsmem = smem;
        // Utilizing warp shuffle operation to reduce warp divergence
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = smem[0];
}