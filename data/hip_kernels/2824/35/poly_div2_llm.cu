#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poly_div2(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        // Store the repeated expressions in temporary variables
        float x = poli[idx];
        float x2 = x * x;
        float x3 = x2 * x;
        float x4 = x3 * x;
        
        // Simplify polynomial evaluation using the temporary variables
        poli[idx] = 5 + x * (7 - x * (9 + x * (5 + x4 + x4))) + x * 0.2;
    }
}