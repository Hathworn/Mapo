#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum4M(float *A, float *B, float *C, const int N) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check boundary condition up front to avoid unnecessary computations
    if (i >= N) return;

    // Cache A[i] and B[i] to reduce memory access
    float ai = A[i];
    float bi = B[i];
    
    // Optimize unrolled loop and arithmetic operations
    #pragma unroll
    for (int j = 0; j < 4; j++) {
        float x = ai / 7.0 * ai;
        C[i] += ai / 3.0 + 20 * bi - ai * x + x * bi * 7;
        i += blockDim.x * gridDim.x;
        
        // Repeat boundary check within loop
        if (i >= N) return;
    }
}