#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *F, double *D)
{
    // Use thread 0 of block 0 to initialize F and D
    if (blockIdx.x == 0 && threadIdx.x == 0)
    {
        *F = 12.1f;  // Use the correct type suffix for float
        *D = 12.1;   // Double type assignment is fine as is
    }
}