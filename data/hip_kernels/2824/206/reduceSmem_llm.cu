#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized using shared memory access and less synchronization
__global__ void reduceSmem(int *g_idata, int *g_odata, unsigned int n) {
    __shared__ int smem[DIM];

    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        int *idata = g_idata + blockIdx.x * blockDim.x;
        smem[tid] = idata[tid];
        __syncthreads();

        if (blockDim.x >= 1024 && tid < 512) smem[tid] += smem[tid + 512];
        __syncthreads();
        if (blockDim.x >= 512 && tid < 256) smem[tid] += smem[tid + 256];
        __syncthreads();
        if (blockDim.x >= 256 && tid < 128) smem[tid] += smem[tid + 128];
        __syncthreads();
        if (blockDim.x >= 128 && tid < 64) smem[tid] += smem[tid + 64];
        __syncthreads();

        // Use warp shuffle for final accumulation within the warp
        if (tid < 32) {
            for (int offset = 32; offset > 0; offset >>= 1) {
                smem[tid] += __shfl_down(smem[tid], offset);
            }
        }

        if (tid == 0) g_odata[blockIdx.x] = smem[0];
    }
}