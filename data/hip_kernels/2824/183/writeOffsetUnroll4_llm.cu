#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void writeOffsetUnroll4(float *A, float *B, float *C, const int n, int offset)
{
    unsigned int i = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // Use a register to avoid recalculating blockDim.x * 4
    int stride = blockDim.x * 4;
    
    unsigned int k = i + offset;
    
    // Ensure correct bounds checking and unrolling
    if (k + 3 * blockDim.x < n)
    {
        C[k]              = A[i]                 + B[i];
        C[k + blockDim.x] = A[i + blockDim.x]    + B[i + blockDim.x];
        C[k + 2 * blockDim.x] = A[i + 2 * blockDim.x] + B[i + 2 * blockDim.x];
        C[k + 3 * blockDim.x] = A[i + 3 * blockDim.x] + B[i + 3 * blockDim.x];
    }
    else 
    {
        for (int j = 0; j < stride && (k + j < n); j += blockDim.x) 
        {
            C[k + j] = A[i + j] + B[i + j];
        }
    }
}