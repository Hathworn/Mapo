#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencilConst2(float *src, float *dst, int size, int raio)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += raio + 1;
    if (idx >= size)
        return;
    
    float out = 0;
    // Use shared memory for stencil weights to reduce global memory accesses
    __shared__ float shared_stencilWeight[2 * raio];
    if (threadIdx.x < 2 * raio) {
        shared_stencilWeight[threadIdx.x] = const_stencilWeight[threadIdx.x];
    }
    __syncthreads();

    // Calculate stencil with unrolled loop and shared memory
    #pragma unroll
    for (int i = -raio; i < raio; i++) {
        out += src[idx + i] * shared_stencilWeight[i + raio];
    }
    dst[idx] = out;
}