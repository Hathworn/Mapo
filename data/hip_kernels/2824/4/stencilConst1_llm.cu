#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void stencilConst1(float *src, float *dst, int size, int raio)
{
    // Calculate unique global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    idx += raio + 1;
    
    // Check if index is within bounds
    if (idx >= size)
        return;
    
    float out = 0.0f;

    // Loop unrolling for performance
    #pragma unroll
    for(int i = -raio; i < raio; i++)
    {
        // Optimize memory access by local variable
        float weight = const_stencilWeight[i + raio];
        out += src[idx + i] * weight;
    }
    
    // Store result in destination array
    dst[idx] = out;
}