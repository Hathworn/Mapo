#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduceUnrolling4 (int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;

    // convert global data pointer to the local pointer of this block
    int *idata = g_idata + blockIdx.x * blockDim.x * 4;

    // unrolling 4
    int sum = 0;
    if (idx + 3 * blockDim.x < n)
    {
        // Load and reduce directly into local variable to avoid global memory writes
        sum += g_idata[idx];
        sum += g_idata[idx + blockDim.x];
        sum += g_idata[idx + 2 * blockDim.x];
        sum += g_idata[idx + 3 * blockDim.x];
    }

    // Write partial sum to idata array after bounds check
    idata[tid] = sum;

    __syncthreads();

    // in-place reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1)
    {
        if (tid < stride)
        {
            idata[tid] += idata[tid + stride];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}