#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N)
{
    // Optimize thread index calculation and avoid redundant operations
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < N) {
        // Directly store result in C without using temporary array
        C[i] = A[i] + B[i];
    }
}