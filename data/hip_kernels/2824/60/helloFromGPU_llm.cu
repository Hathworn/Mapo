#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU() {
    // Calculate global thread index for this kernel
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Print message only for the first thread for efficiency
    if (idx == 0) {
        printf("Hello World from GPU!\n");
    }
}