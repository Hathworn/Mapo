#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void poli2(float* poli, const int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index is in bounds before processing
    if (idx < N) {
        float x = poli[idx];
        float x2 = x * x; // Precompute x^2 for reuse
        poli[idx] = (4 * x * x2) + (3 * x2) - (7 * x) + 5; // Simplified polynomial calculation
    }
}