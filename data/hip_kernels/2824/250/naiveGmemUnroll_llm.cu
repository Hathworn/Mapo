#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void naiveGmemUnroll(float *out, float *in, const int nx, const int ny)
{
    // Calculate global indices with unrolling
    unsigned int ix = 2 * blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Calculate linear indices for input and output
    unsigned int ti = iy * nx + ix;
    unsigned int to = ix * ny + iy;

    // Optimize boundary checks to prevent redundant computations
    if (iy < ny && ix + blockDim.x < nx)
    {
        // Unrolled assignments for adjacent elements
        out[to]                  = in[ti];
        out[to + ny * blockDim.x] = in[ti + blockDim.x];
    }
}