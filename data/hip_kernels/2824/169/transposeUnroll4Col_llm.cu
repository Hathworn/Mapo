#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeUnroll4Col(float *out, float *in, const int nx, const int ny)
{
    // Compute global thread indices
    unsigned int ix = blockDim.x * blockIdx.x * 4 + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Compute input and output index
    unsigned int ti = iy * nx + ix; // access in rows
    unsigned int to = ix * ny + iy; // access in columns

    // Optimize memory access with coalesced reads and unroll
    if (ix + 3 * blockDim.x < nx && iy < ny)
    {
        float in1 = in[to];
        float in2 = in[to + blockDim.x * ny];
        float in3 = in[to + 2 * blockDim.x * ny];
        float in4 = in[to + 3 * blockDim.x * ny];

        out[ti] = in1;
        out[ti + blockDim.x] = in2;
        out[ti + 2 * blockDim.x] = in3;
        out[ti + 3 * blockDim.x] = in4;
    }
}