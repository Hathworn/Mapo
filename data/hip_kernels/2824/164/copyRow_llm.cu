#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copyRow(float *out, float *in, const int nx, const int ny)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure the index is within bounds before copying
    if (ix < nx && iy < ny) 
    {
        out[iy * nx + ix] = in[iy * nx + ix];
    }
}