#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void transposeDiagonalRow(float *out, float *in, const int nx, const int ny)
{
    // Calculate the coordinates using grid-stride loop to handle larger arrays
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int iy = blockDim.y * blockIdx.y + threadIdx.y;

    // Loop to ensure all elements are processed even if data size exceeds grid size
    for (unsigned int i = ix; i < nx; i += blockDim.x * gridDim.x)
    {
        for (unsigned int j = iy; j < ny; j += blockDim.y * gridDim.y)
        {
            // Ensure bounds are respected and perform the transpose
            if (i < nx && j < ny)
            {
                out[i * ny + j] = in[j * nx + i];
            }
        }
    }
}