#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sumMatrixGPU(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate global index
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Combine index calculations
    unsigned int idx = iy * nx + ix;

    // Use single condition check
    if (idx < nx * ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];  // Perform element-wise addition
    }
}