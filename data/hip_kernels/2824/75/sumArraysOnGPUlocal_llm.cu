#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPUlocal(float *A, float *B, float *C, const int N)
{
    // Use register local variable instead of array
    float local;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        // Calculate only the needed local value
        local = 2 * A[i];
        C[i] = A[i] + B[i] + local;
    }
}