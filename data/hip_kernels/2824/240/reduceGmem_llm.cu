#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reduceGmem(int *g_idata, int *g_odata, unsigned int n)
{
    // set thread ID
    unsigned int tid = threadIdx.x;
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int *idata = g_idata + blockIdx.x * blockDim.x;

    // boundary check
    if (idx >= n) return;

    // in-place reduction in global memory with loop unrolling
    #pragma unroll
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
    {
        if (tid < s)
        {
            idata[tid] += idata[tid + s];
        }
        __syncthreads();
    }

    // unrolling warp
    if (tid < 32)
    {
        volatile int *vsmem = idata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid +  8];
        vsmem[tid] += vsmem[tid +  4];
        vsmem[tid] += vsmem[tid +  2];
        vsmem[tid] += vsmem[tid +  1];
    }

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = idata[0];
}