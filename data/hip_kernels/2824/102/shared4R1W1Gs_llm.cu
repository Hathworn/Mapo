#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void shared4R1W1Gs(float *A, float *B, float *C, const int N)
{
    __shared__ float Smem[512];

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input to shared memory with bounds check
    if (i < N) {
        Smem[threadIdx.x] = A[i];
    }
    __syncthreads();

    // Compute only if within bounds
    if (i < N) {
        C[i] = Smem[threadIdx.x] + Smem[(threadIdx.x + 1) % 512] + Smem[(threadIdx.x + 2) % 512] + Smem[(threadIdx.x + 3) % 512] + Smem[(threadIdx.x + 4) % 512];
    }
}