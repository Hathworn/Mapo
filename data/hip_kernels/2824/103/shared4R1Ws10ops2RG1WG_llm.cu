#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void shared4R1Ws10ops2RG1WG(float *A, float *B, float *C, const int N)
{
    __shared__ float Smem[512];

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory with bounds check
    if (i < N) {
        Smem[threadIdx.x] = A[i];
    }
    __syncthreads();

    // Use a single conditional for computation
    if (i < N) {
        float a_i = A[i];
        float b_i = B[i];
        
        // Compute reused expressions once
        float expr1 = a_i * a_i;
        float expr2 = a_i * b_i;
        float expr3 = b_i * b_i;
        
        // Streamline the expression
        C[i] = a_i + b_i - expr1 + 3 * b_i - 4 * expr2 + expr3 * 7 - 8 
               + Smem[(threadIdx.x + 1) % 512] 
               + Smem[(threadIdx.x + 2) % 512] 
               + Smem[(threadIdx.x + 3) % 512] 
               + Smem[(threadIdx.x + 4) % 512];
    }
}