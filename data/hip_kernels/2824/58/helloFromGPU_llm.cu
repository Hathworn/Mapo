#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU()
{
    // Use only the first thread of the first block for printf
    if ( threadIdx.x == 0 && blockIdx.x == 0 ) {
        printf("Hello World from GPU! %d\n", blockIdx.x);
    }
}