#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void coalesced(float *A, float *C, const int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for coalesced access
    __shared__ float sharedA[BLOCK_SIZE];
    
    if (i < N) {
        // Load data into shared memory
        sharedA[threadIdx.x] = A[i];
        __syncthreads(); // Ensure all loads are complete
        
        // Store from shared memory to global memory
        C[i] = sharedA[threadIdx.x];
    }
}
```
