#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C, const int N) {
    // Calculate global thread ID using block & thread indices
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the thread ID is within bounds before accessing memory
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}