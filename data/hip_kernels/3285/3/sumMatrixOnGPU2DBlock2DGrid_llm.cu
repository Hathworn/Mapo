#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel for matrix summation
__global__ void sumMatrixOnGPU2DBlock2DGrid(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    // Calculate global thread index for x and y dimensions
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    // Compute 1D index from 2D coordinates
    unsigned int idx = iy * nx + ix;

    // Check boundaries and perform addition only if within bounds
    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}