#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void helloFromGPU() {
    // Use thread and block indices to distribute work if needed
    if (threadIdx.x == 0 && blockIdx.x == 0) {  // Optimize by printing from only one thread
        printf("Hello from GPU!\n");
    }
}