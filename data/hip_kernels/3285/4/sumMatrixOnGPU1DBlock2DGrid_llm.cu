#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumMatrixOnGPU1DBlock2DGrid(float *MatA, float *MatB, float *MatC, int nx, int ny)
{
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * nx + ix;

    // Ensure threads operate within matrix bounds
    if (ix < nx && iy < ny)
    {
        MatC[idx] = MatA[idx] + MatB[idx];
    }
}