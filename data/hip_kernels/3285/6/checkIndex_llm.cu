#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void checkIndex() {
    // Cache frequently accessed values in registers to reduce latency
    int bx = blockIdx.x; 
    int by = blockIdx.y; 
    int bz = blockIdx.z;
    int tx = threadIdx.x; 
    int ty = threadIdx.y; 
    int tz = threadIdx.z;
    int bdx = blockDim.x; 
    int bdy = blockDim.y; 
    int bdz = blockDim.z;
    int gdx = gridDim.x; 
    int gdy = gridDim.y; 
    int gdz = gridDim.z;

    printf("blockIdx:(%d, %d, %d) threadIdx:(%d, %d, %d) blockDim:(%d, %d, %d) gridDim:(%d, %d, %d)\n",
           bx, by, bz, tx, ty, tz, bdx, bdy, bdz, gdx, gdy, gdz);
}