#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void printThreadIndex(int *A, const int nx, const int ny)
{
    // Optimize by checking if thread index is within bounds before proceeding
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;

    if (ix < nx && iy < ny) { // Ensure index is within the array bounds
        unsigned int idx = iy * nx + ix;
        printf("thread_id (%d,%d) block_id (%d,%d) coordinate (%d,%d) global index"
        " %2d ival %2d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y,
        ix, iy, idx, A[idx]);
    }
}