#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
int addWithCuda(int *c, const int *a, const int *b, unsigned int size);
bool isLoadDevice = false;

// Optimized kernel for better parallelism and memory access
__global__ void addKernel(int *c, const int *a, const int *b, int n) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) // Ensure threads do not access out-of-bound memory
    {
        c[i] = a[i] + b[i];
    }
}