#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initSquare(float* a, float* x, float totalX, int n, int ghosts) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int index;
    
    // Unrolling the loop for better performance
    for (int j = 0; (index = blockDim.x * j + i) < n; j++) {
        if (index > n / 3 && index < 2 * n / 3)
            a[index + ghosts] = 1.5f;
        else
            a[index + ghosts] = 0.5f;
    }
    
    if (i == 0) { // Only one thread handles the boundary conditions
        for (int j = 0; j < ghosts; j++) {
            a[j] = a[j + n];
            a[n + ghosts + j] = a[ghosts + j];
            // Removing commented out code for clarity
        }
    }
}