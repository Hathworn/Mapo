#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initialize(float* a, float* oA, float* x, float totalSize, int n, int ghosts) {
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    int stride = gridDim.x * blockDim.x; // Calculate stride for grid parallelism
    for(int i = index; i < n + 2 * ghosts; i += stride) { // Use stride loop for better performance
        a[i] = 0;
        oA[i] = 0;
        x[i] = totalSize / n;
    }
}