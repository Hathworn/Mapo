#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void initSinusoid(float* a, float* x, float totalX, int n, int ghosts, float shift, float amp) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    // Use shared memory to store intermediate results for better performance
    extern __shared__ float sharedTemp[];
    
    float temp = 0.0f;
    // Ensure threads do not perform unnecessary calculations
    int gridStride = blockDim.x * gridDim.x;
    
    for (int index = i; index < n; index += gridStride) {
        temp = 0.0f;
        for (int z = 0; z < index; z++) {
            temp += x[z + ghosts];
        }
        sharedTemp[threadIdx.x] = temp;
        
        __syncthreads();
        
        // Calculate the sinusoidal value using shared memory
        a[index + ghosts] = sinpi((sharedTemp[threadIdx.x] / totalX) * 2) * amp + shift;
    }
    
    // Manage boundary conditions synchronously
    if (i < ghosts) {
        a[i] = a[i + n];
        a[n + ghosts + i] = a[ghosts + i];
    }
    __syncthreads();
}