#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float colellaEvenInterp(float* in) {
    return (7.0/12)*(*(in+1) - *(in-1)) - (1.0/12)*((*(in+2) + *(in-1)) - (*(in+1) + *(in-2)));
}

__device__ float linInterp(float* in) { // Dangerous function, ensure it's used only on in-bound parts of array
    return ((*(in+1) + *in)/2) - ((*in + *(in-1))/2);
}

__global__ void advect(float* a, float* oA, float* x, float u, int n, int ghosts, float tmax) {
    extern __shared__ float sharedData[]; // Use dynamic shared memory
    float& dt = sharedData[0];
    float& minDx = sharedData[1];
    float& timeElapsed = sharedData[2];
    int& counter = reinterpret_cast<int&>(sharedData[3]);

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    timeElapsed = 0;

    if (i == 0) {
        minDx = x[0];
        dt = (minDx / u) / 500;
        counter = 0;
    }
    __syncthreads();

    while (timeElapsed < tmax) {
        for (int j = 0; blockDim.x * j + i < n; j++) {
            int index = j * blockDim.x + i + ghosts;
            oA[index] = a[index] - dt * u * colellaEvenInterp(&a[index]) / x[index];
            a[index] = oA[index];
        }

        __syncthreads();

        if (counter == 0) {
            for (int j = 0; blockDim.x * j + i < n; j++) {
                int index = j * blockDim.x + i + ghosts;
                printf("%10f\t%10d\t%f\n", timeElapsed, index - ghosts, a[index]);
            }
        }

        if (i == 0) { // Copy over for boundary conditions
            for (int j = 0; j < ghosts; j++) {
                a[j] = a[j + n];
                a[n + ghosts + j] = a[ghosts + j];
            }
            timeElapsed += dt;
            counter++;
            if (counter == 10000) counter = 0;
        }
    }
}