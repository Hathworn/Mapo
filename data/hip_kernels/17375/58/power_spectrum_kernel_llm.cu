#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global thread index
    const float *Ar = A_in + blockIdx.x * ldi;
    float *Aw = A_out + blockIdx.x * ldo;

    int half_length = row_length / 2;
    
    if (thread_id < half_length) {
        // Handle special case: idx == 0
        if (thread_id == 0) {
            float real = Ar[0];
            float im = Ar[row_length];

            if (use_power) {
                Aw[0] = real * real;
                Aw[half_length] = im * im;
            } else {
                Aw[0] = fabs(real);
                Aw[half_length] = fabs(im);
            }
        } 
        // General case for idx != 0
        else {
            float2 val = reinterpret_cast<const float2 *>(Ar)[thread_id];
            float ret = val.x * val.x + val.y * val.y;
            if (use_power) {
                Aw[thread_id] = ret;
            } else {
                Aw[thread_id] = sqrtf(ret);
            }
        }
    }
}