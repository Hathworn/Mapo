#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void apply_lifter_and_floor_energy( int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    int thread_id = threadIdx.x;
    int frame = blockIdx.x;
    
    float *feats = features + frame * ldf;
    
    // Use shared memory for lifter coefficients
    __shared__ float shared_lifter_coeffs[CU1DBLOCK];
    if (thread_id < num_cols) {
        shared_lifter_coeffs[thread_id] = lifter_coeffs[thread_id];
    }
    __syncthreads();

    // Apply lifter coefficients in parallel
    if (cepstral_lifter != 0.0f) {
        for (int c = thread_id; c < num_cols; c += CU1DBLOCK) {
            feats[c] *= shared_lifter_coeffs[c];
        }
    }

    // Only apply energy in first thread of each block
    if (use_energy && thread_id == 0) {
        float energy = log_energy[frame];
        float log_energy_floor = logf(energy_floor);

        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        feats[0] = energy;
    }
}