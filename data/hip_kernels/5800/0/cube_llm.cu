#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cube(float *d_out, float *d_in) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global index
    float f = d_in[idx];
    d_out[idx] = f * f * f; // Cube calculation remains the same
}