#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Use the block index and thread index for a more scalable approach
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global index
    if (i < blockDim.x) { // Check bounds to avoid out-of-bounds access
        c[i] = a[i] + b[i];
        // Removed unnecessary printf for cleaner execution
    }
}