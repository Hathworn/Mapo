#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void squareMatrixMulKernel(int *c, int *a, int *b, int arrayWidth)
{
    // Sum initialized to zero
    float sum = 0.0f;

    // Calculate global thread indices for row and column
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure indices are within matrix bounds
    if (row < arrayWidth && col < arrayWidth) {
        // Perform dot product of row and column
        for (int i = 0; i < arrayWidth; ++i) {
            float Aelement = static_cast<float>(a[row * arrayWidth + i]);
            float Belement = static_cast<float>(b[i * arrayWidth + col]);
            sum += Aelement * Belement;
        }
        // Store the result in matrix c
        c[row * arrayWidth + col] = static_cast<int>(sum);
    }
}