#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define TILE_SIZE 16

__global__ void squareMatrixMulKernel(int *c, int *a, int *b, int arrayWidth)
{
    // Allocate shared memory for tiles of input matrices
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];

    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;

    float sum = 0;

    // Iterate over tiles
    for (int tileIdx = 0; tileIdx < arrayWidth/TILE_SIZE; ++tileIdx) {
        // Load data into shared memory for the current tile
        tileA[threadIdx.y][threadIdx.x] = a[row * arrayWidth + tileIdx * TILE_SIZE + threadIdx.x];
        tileB[threadIdx.y][threadIdx.x] = b[(tileIdx * TILE_SIZE + threadIdx.y) * arrayWidth + col];

        // Synchronize threads to ensure all data is loaded
        __syncthreads();

        // Perform tile matrix multiplication
        for (int i = 0; i < TILE_SIZE; ++i) {
            sum += tileA[threadIdx.y][i] * tileB[i][threadIdx.x];
        }

        // Synchronize threads before loading next tile
        __syncthreads();
    }

    // Write the result into global memory
    c[row * arrayWidth + col] = sum;
}