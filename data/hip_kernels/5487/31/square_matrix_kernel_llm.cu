#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    int i = blockIdx.y * blockDim.y + threadIdx.y; // compute row index
    int j = blockIdx.x * blockDim.x + threadIdx.x; // compute column index

    // Ensure i and j are within bounds
    if (i < num_rows && j < num_cols) {
        float f = feats[i * ldf + j];  // load feature value
        feats_sq[i * lds + j] = f * f; // store squared value
    }
}