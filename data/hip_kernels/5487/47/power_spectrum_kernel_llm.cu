#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
    float *Ar = A_in + blockIdx.x * ldi;
    float *Aw = A_out + blockIdx.x * ldo;
    int half_length = row_length / 2;

    // Process non-special cases
    if (thread_id < half_length && thread_id != 0) {
        float2 val = reinterpret_cast<float2 *>(Ar)[thread_id];
        float ret = val.x * val.x + val.y * val.y;
        Aw[thread_id] = use_power ? ret : sqrtf(ret);
    }

    // Handle special case in first thread of the block
    if (threadIdx.x == 0) {
        float real = Ar[0];
        float im = Ar[row_length];

        if (use_power) {
            Aw[0] = real * real;
            Aw[half_length] = im * im;
        } else {
            Aw[0] = fabs(real);
            Aw[half_length] = fabs(im);
        }
    }
}