#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_lifter_and_floor_energy(int num_frames, int num_cols, float cepstral_lifter, bool use_energy, float energy_floor, float *log_energy, float *lifter_coeffs, float *features, int32_t ldf) {
    // Calculate global thread index
    int global_thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Apply lifter coefficients using all threads
    if (cepstral_lifter != 0.0f) {
        if (global_thread_id < num_frames * num_cols) {
            int frame = global_thread_id / num_cols;
            int c = global_thread_id % num_cols;
            float *feats = features + frame * ldf;
            feats[c] *= lifter_coeffs[c];
        }
    }
    
    // Use first thread to apply energy per frame
    if (use_energy && threadIdx.x == 0) {
        float energy = log_energy[blockIdx.x];
        float log_energy_floor = log(energy_floor);
        
        if (energy_floor > 0.0f && energy < log_energy_floor) {
            energy = log_energy_floor;
        }
        features[blockIdx.x * ldf] = energy;
    }
}