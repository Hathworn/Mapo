#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Calculate global index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we only modify the intended element when using multiple threads
    if (idx == 0) {
        *accuracy /= N;
    }
}