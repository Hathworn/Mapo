#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int GPUKernel_Position(int i, int j) {
    // Simplified conditional logic for position calculation
    return (i < j) ? j * (j + 1) / 2 + i : i * (i + 1) / 2 + j;
}

__global__ void GPUKernel_Vp(int a, int v, double* in, double* out) {
    // Calculate global thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Exit early if id is out of valid range
    if (id >= v * v * v) return;

    // Simplify variable calculations using integer division and modulus directly
    int d = id % v;
    int b = (id / v) % v;
    int c = id / (v * v);

    // Additional conditions to filter out invalid combinations
    if (b < a || d > c) return;

    // Calculate new indices based on position function
    int cd = GPUKernel_Position(c, d);
    int vtri = v * (v + 1) / 2;

    // Compute output based on in and calculated indices
    out[(b - a) * vtri + cd] = in[(b - a) * v * v + d * v + c] + in[(b - a) * v * v + c * v + d];
}