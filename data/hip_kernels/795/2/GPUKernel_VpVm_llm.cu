#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int GPUKernel_Position(int i, int j) {
    if (i < j) {
        return j * (j + 1) / 2 + i;
    }
    return i * (i + 1) / 2 + j;
}

__global__ void GPUKernel_VpVm(int a, int v, double* in, double* outp, double* outm) {
    int blockid = blockIdx.x * gridDim.y + blockIdx.y;
    int id = blockid * blockDim.x + threadIdx.x;

    int v2 = v * v;

    // Use early exit to minimize work
    if (id >= v2 * v) return;

    int d = id % v;
    int b = (id - d) % (v2) / v;

    if (b < a) return;

    int bma = b - a;

    int c = (id - d - b * v) / (v2);

    if (d > c) return;

    int cd = GPUKernel_Position(c, d);
    int vtri = v * (v + 1) / 2;

    // Use temporary variables to avoid redundant calculations
    double in1 = in[bma * v2 + d * v + c];
    double in2 = in[bma * v2 + c * v + d];

    // Compute outputs directly
    outp[bma * vtri + cd] = in1 + in2;
    outm[bma * vtri + cd] = in1 - in2;
}