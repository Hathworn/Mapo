#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int GPUKernel_Position(int i, int j) {
    return (i < j) ? j * (j + 1) / 2 + i : i * (i + 1) / 2 + j;
}

__global__ void GPUKernel_Vm(int a, int v, double* in, double* out) {

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit for threads outside valid range
    if (id >= (v * v * v)) return;
    
    int v2 = v * v; // Precompute to avoid recalculating
    int d = id % v;
    int b = (id / v) % v;
    int c = id / v2;

    if (b < a || d > c) return;

    int cd = GPUKernel_Position(c, d);
    int vtri = v * (v + 1) / 2;
    int offset = (b - a) * v2;
    
    out[(b - a) * vtri + cd] = in[offset + d * v + c] - in[offset + c * v + d];
}