#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int GPUKernel_Position(int i, int j) {
    if (i < j) {
        return j * (j + 1) / 2 + i;
    }
    return i * (i + 1) / 2 + j;
}

__global__ void GPUKernel_VpVm_v2(int a, int b, int v, double *in, double *outp, double *outm) {
    int blockid = blockIdx.x * gridDim.y + blockIdx.y;
    int id = blockid * blockDim.x + threadIdx.x;
    int v2 = v * v;

    // Use shared memory to avoid repeated global memory reads
    __shared__ double s_in[1024];  // Adjust size based on actual needs
    if (id < v2) {
        s_in[threadIdx.x] = in[id];
    }
    __syncthreads();

    if (id >= v2) return;

    int d = id % v;
    int c = id / v;

    if (d > c) return;

    int cd = GPUKernel_Position(c, d);

    // Use shared memory for intermediate calculations
    outp[cd] = s_in[d * v + c] + s_in[c * v + d];
    outm[cd] = s_in[d * v + c] - s_in[c * v + d];
}