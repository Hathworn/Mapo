#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GPUKernel_Iqdb(int a, int v, int nQ, double *in, double *out) {

    // Optimized id calculation
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id >= v * v * nQ) return;

    int q = id % nQ;
    int d = (id / nQ) % v;
    int b = id / (nQ * v);

    if (b < a) return;

    int id2 = (b - a) * nQ * v + d * nQ + q;
    out[id2] = in[id];
}