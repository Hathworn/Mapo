#include "hip/hip_runtime.h"
```c++
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void GPUKernel_VpVm_tiled(int a, int bstart, int bsize,int v,double * in,double * outp,double * outm) {

    int blockid = blockIdx.x * gridDim.y + blockIdx.y;
    int id      = blockid * blockDim.x + threadIdx.x;

    int v2 = v * v;

    if (id >= v2 * bsize) return; // Boundary condition

    // id: b * v2 + c * v + d
    int d = id % v;
    int c = (id / v) % v;
    int b = id / (v * v);

    if (d > c) return; // Skip the unnecessary computations

    if (b + bstart < a) return;

    int cd   = c * (c + 1) / 2 + d; // Map (c, d) to 1D array
    int vtri = v * (v + 1) / 2;
    int bv2  = b * v2;

    outp[b * vtri + cd] = in[bv2 + d * v + c] + in[id]; // Compute output for outp
    outm[b * vtri + cd] = in[bv2 + d * v + c] - in[id]; // Compute output for outm
}