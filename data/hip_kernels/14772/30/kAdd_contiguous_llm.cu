#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x; // Compute stride for better readability

    // Unrolling loop to improve performance
    for (unsigned int i = idx; i < numEls; i += stride * 4) {
        if (i < numEls) dest[i] = a[i] + b[i];
        if (i + stride < numEls) dest[i + stride] = a[i + stride] + b[i + stride];
        if (i + 2 * stride < numEls) dest[i + 2 * stride] = a[i + 2 * stride] + b[i + 2 * stride];
        if (i + 3 * stride < numEls) dest[i + 3 * stride] = a[i + 3 * stride] + b[i + 3 * stride];
    }
}