#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_scatter_cu_z(double *c, const double *pwcc, const double scale, const int ngpts, const int nmaps, const int *ghatmap) {
    int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    if (igpt < ngpts) {
        int index = 2 * ghatmap[igpt];
        c[index] = scale * pwcc[2 * igpt];
        c[index + 1] = scale * pwcc[2 * igpt + 1];
        if (nmaps == 2) {
            int index2 = 2 * ghatmap[igpt + ngpts];
            c[index2] = scale * pwcc[2 * igpt];
            c[index2 + 1] = -scale * pwcc[2 * igpt + 1];
        }
    }
}