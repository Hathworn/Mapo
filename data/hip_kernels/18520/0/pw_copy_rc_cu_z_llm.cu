
#include <hip/hip_runtime.h>
__global__ void pw_copy_rc_cu_z(const double *din, double *zout, const int n) {
    const int igpt = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    if (igpt < n) {
        int index = 2 * igpt;
        zout[index] = din[igpt];
        zout[index + 1] = 0.0;
    }
}