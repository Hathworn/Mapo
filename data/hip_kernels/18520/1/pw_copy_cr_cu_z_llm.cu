#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_copy_cr_cu_z(const double *zin, double *dout, const int n)
{
    int i = blockIdx.x + blockIdx.y * gridDim.x;
    i = i * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x * gridDim.y;
    for (; i < n; i += stride)
    {
        dout[i] = zin[2 * i];
    }
}