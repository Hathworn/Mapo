#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void pw_gather_cu_z(double *pwcc, const double *c, const double scale, const int ngpts, const int *ghatmap) 
{
    const int igpt = (gridDim.x * blockIdx.y + blockIdx.x) * blockDim.x + threadIdx.x;
    if (igpt < ngpts)
    {
        const int idx = ghatmap[igpt];
        double2 val = reinterpret_cast<const double2*>(c)[idx];
        val.x *= scale;
        val.y *= scale;
        reinterpret_cast<double2*>(pwcc)[igpt] = val;
    }
}