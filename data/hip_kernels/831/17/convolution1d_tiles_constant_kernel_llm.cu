#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution1d_tiles_constant_kernel(int *In, int *Out) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x; // Index 1d iterator.
    __shared__ int Tile[TILE_SIZE + Mask_size - 1];
    int n = Mask_size / 2;

    // Load left halo elements into shared memory
    int halo_left_index = index - n;
    if (threadIdx.x >= blockDim.x - n) {
        Tile[threadIdx.x - (blockDim.x - n)] = (halo_left_index < 0) ? 0 : In[halo_left_index];
    }

    // Load centre elements into shared memory
    Tile[n + threadIdx.x] = (index < N_elements) ? In[index] : 0;

    // Load right halo elements into shared memory
    int halo_right_index = index + blockDim.x;
    if (threadIdx.x < n) {
        Tile[n + blockDim.x + threadIdx.x] = (halo_right_index >= N_elements) ? 0 : In[halo_right_index];
    }

    __syncthreads();

    // Compute convolution using shared memory
    int Value = 0;
    for (unsigned int j = 0; j < Mask_size; j++) {
        Value += Tile[threadIdx.x + j] * Global_Mask[j];
    }

    // Store the result
    if (index < N_elements) {
        Out[index] = Value;
    }
}