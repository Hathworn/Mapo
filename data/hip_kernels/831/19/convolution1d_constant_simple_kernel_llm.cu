#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void convolution1d_constant_simple_kernel(int *In, int *Out){

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ float N_ds[TILE_SIZE];
    N_ds[threadIdx.x] = (i < N_elements) ? In[i] : 0; // Load handling boundary
    __syncthreads();

    int N_start_point = i - (Mask_size / 2);
    int Pvalue = 0;
    
    // Unroll loop for better performance
    #pragma unroll
    for (int j = 0; j < Mask_size; j++) {
        int N_index = N_start_point + j;
        if (N_index >= 0 && N_index < N_elements) {
            Pvalue += ((blockIdx.x * blockDim.x <= N_index) && (N_index < (blockIdx.x + 1) * blockDim.x))
                      ? N_ds[threadIdx.x + j - (Mask_size / 2)] * Global_Mask[j]
                      : In[N_index] * Global_Mask[j];
        }
    }
    if (i < N_elements) { // Writing result only for valid indices
        Out[i] = Pvalue;
    }
}