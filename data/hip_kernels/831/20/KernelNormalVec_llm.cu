#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KernelNormalVec(double *g_idata, double *g_odata, int l) {

    __shared__ double sdata[BLOCK_SIZE];

    // Improved boundary checking
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + tid;

    // Efficient conditional loading with fewer checks
    double value = (i < l) ? g_idata[i] : 0.0;
    sdata[tid] = value;

    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Directly write result for this block to global memory
    if (tid == 0) {
        g_odata[blockIdx.x] = sdata[0];
    }
}