#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define FILTER_RADIUS 1
#define BLOCK_WIDTH (TILE_WIDTH + 2 * FILTER_RADIUS)
#define BLOCK_HEIGHT (TILE_HEIGHT + 2 * FILTER_RADIUS)
#define FILTER_AREA ((2 * FILTER_RADIUS + 1) * (2 * FILTER_RADIUS + 1))

__global__ void sobelFilterShared3(unsigned char* g_DataIn, unsigned char *g_DataOut, unsigned int width, unsigned int height) {
    extern __shared__ char sharedMem[];

    int x = blockIdx.x * TILE_WIDTH + threadIdx.x - FILTER_RADIUS;
    int y = blockIdx.y * TILE_HEIGHT + threadIdx.y - FILTER_RADIUS;

    // Clamp to the center
    x = max(FILTER_RADIUS, min(x, width - FILTER_RADIUS - 1));
    y = max(FILTER_RADIUS, min(y, height - FILTER_RADIUS - 1));

    unsigned int index = y * width + x;
    unsigned int sharedIndex = threadIdx.y * blockDim.x + threadIdx.x;

    // Load data into shared memory
    sharedMem[sharedIndex] = g_DataIn[index];
    __syncthreads();

    if(    threadIdx.x >= FILTER_RADIUS && threadIdx.x < blockDim.x - FILTER_RADIUS
        && threadIdx.y >= FILTER_RADIUS && threadIdx.y < blockDim.y - FILTER_RADIUS) {
        
        int sum = 0;
        for(int dy = -FILTER_RADIUS; dy <= FILTER_RADIUS; ++dy) {
            for(int dx = -FILTER_RADIUS; dx <= FILTER_RADIUS; ++dx) {
                // Calculate offset in shared memory
                int sharedOffset = (threadIdx.y + dy) * blockDim.x + (threadIdx.x + dx);
                sum += (int)sharedMem[sharedOffset];
            }
        }
        // Write result to global memory
        g_DataOut[index] = (unsigned char)(sum / FILTER_AREA);
    }
}