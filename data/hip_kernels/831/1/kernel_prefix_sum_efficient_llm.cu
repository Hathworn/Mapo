#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#define SIZE 60 // Global Size
#define BLOCK_SIZE 1024
using namespace std;

// :::: Kernel
__global__ void kernel_prefix_sum_efficient(double *g_idata, double *g_odata, int l) {
    // Shared memory allocation for block-wide data
    __shared__ double temp[BLOCK_SIZE];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;

    // Load input into shared memory
    if (index < l) {
        temp[tid] = g_idata[index];
    } else {
        temp[tid] = 0; // Avoid out-of-bounds access
    }
    __syncthreads();

    // Sequential addressing for prefix sum
    for (int offset = 1; offset < blockDim.x; offset *= 2) {
        if (tid >= offset) {
            temp[tid] += temp[tid - offset];
        }
        __syncthreads(); // Ensure all additions are done before the next step
    }

    // Write result to g_odata
    if (index < l) {
        g_odata[index] = temp[tid];
    }
}