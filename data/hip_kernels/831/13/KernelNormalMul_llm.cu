#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KernelNormalMul(int *Mat1, int *Mat2, int *Mat3, int m, int n, int p) {
    int j = threadIdx.y + blockDim.y * blockIdx.y; // row
    int i = threadIdx.x + blockDim.x * blockIdx.x; // col

    // Ensure valid thread indices before proceeding
    if (j < m && i < p) {
        int value = 0;
        // Unroll loop for better memory access and coalescing
        for (int k = 0; k < n; k += 4) {
            if (k < n) value += Mat1[n * j + k] * Mat2[p * k + i];
            if (k + 1 < n) value += Mat1[n * j + (k + 1)] * Mat2[p * (k + 1) + i];
            if (k + 2 < n) value += Mat1[n * j + (k + 2)] * Mat2[p * (k + 2) + i];
            if (k + 3 < n) value += Mat1[n * j + (k + 3)] * Mat2[p * (k + 3) + i];
        }
        Mat3[p * j + i] = value;
    }
}