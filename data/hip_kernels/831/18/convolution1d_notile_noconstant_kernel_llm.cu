#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolution1d_notile_noconstant_kernel(int *In, int *Out) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x; // Index 1d iterator.
    
    // Early return if index is out of bound
    if (index >= N_elements) return; 
    
    int Value = 0;
    int N_start_point = index - (Mask_size / 2);
    
    for (int j = 0; j < Mask_size; j++) {
        int neighborIndex = N_start_point + j;

        // Load element only if it's within bounds, also hoisting boundary check
        if (neighborIndex >= 0 && neighborIndex < N_elements) {
            Value += In[neighborIndex] * Global_Mask[j];
        }
    }
    
    Out[index] = Value;
}