#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gray(unsigned char *In, unsigned char *Out, int Row, int Col) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if the current thread is within image bounds
    if ((row < Col) && (col < Row)) {
        // Calculate the index once and reuse
        int idx = (row * Row + col) * 3;
        // Apply the grayscale conversion
        Out[row * Row + col] = In[idx + 2] * 0.299f + In[idx + 1] * 0.587f + In[idx] * 0.114f;
    }
}