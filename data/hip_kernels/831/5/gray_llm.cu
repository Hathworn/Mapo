#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gray(unsigned char *In, unsigned char *Out, int Row, int Col) {
    // Compute absolute index based on row and column
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure indices are within bounds
    if ((row < Col) && (col < Row)) {
        int index = row * Row + col;
        int rgbIndex = index * 3;
        
        // Optimize memory accesses by calculating output directly
        unsigned char r = In[rgbIndex];
        unsigned char g = In[rgbIndex + 1];
        unsigned char b = In[rgbIndex + 2];
        
        Out[index] = r * 0.114f + g * 0.587f + b * 0.299f;
    }
}