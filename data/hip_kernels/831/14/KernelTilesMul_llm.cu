#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16

__global__ void KernelTilesMul(int *Mat1, int *Mat2, int *Mat3, int rowM1, int colM1, int colM2) {

    __shared__ int Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ int Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    int Pvalue = 0;

    // Use a single outer loop for shared memory loading and processing
    for (int k = 0; k < (colM1 + TILE_WIDTH - 1) / TILE_WIDTH; ++k) {

        // Load data into shared memory with conditional
        if (k * TILE_WIDTH + tx < colM1 && row < rowM1) {
            Mds[ty][tx] = Mat1[row * colM1 + k * TILE_WIDTH + tx];
        } else {
            Mds[ty][tx] = 0;
        }

        if (k * TILE_WIDTH + ty < colM1 && col < colM2) {
            Nds[ty][tx] = Mat2[(k * TILE_WIDTH + ty) * colM2 + col];
        } else {
            Nds[ty][tx] = 0;
        }

        __syncthreads();

        // Unrolling the loop to improve performance
        #pragma unroll
        for (int n = 0; n < TILE_WIDTH; ++n) {
            Pvalue += Mds[ty][n] * Nds[n][tx];
        }

        __syncthreads();
    }

    // Write the result to matrix C if within boundary
    if (row < rowM1 && col < colM2) {
        Mat3[row * colM2 + col] = Pvalue;
    }
}