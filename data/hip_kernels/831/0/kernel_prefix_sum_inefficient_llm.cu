#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#define SIZE 60 // Global Size
#define BLOCK_SIZE 1024
using namespace std;

// Kernel
__global__ void kernel_prefix_sum_inefficient(double *g_idata, double *g_odata, int l) {
    __shared__ double sdata[BLOCK_SIZE];
    
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load data into shared memory; avoid branch by always using i-1 and setting 0 outside bounds
    sdata[tid] = (i < l && tid != 0) ? g_idata[i - 1] : 0.0; 
    __syncthreads(); // Ensure all data is loaded into shared memory

    // Perform in-place prefix sum using shared memory
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        double val = (tid >= s) ? sdata[tid - s] : 0.0;
        __syncthreads(); // Prevent race conditions
        sdata[tid] += val;
        __syncthreads(); // Ensure correctness for next step
    }

    // Write the result to the output global memory
    if (i < l) {
        g_odata[i] = sdata[tid];
    }
}