#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KernelTilesMul(float *Mat1, float *Mat2, float *Mat3, int rowM1, int colM1, int colM2) {
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row = by * TILE_WIDTH + ty;
    int col = bx * TILE_WIDTH + tx;

    float Pvalue = 0.0;

    for (int k = 0; k < (colM1 + TILE_WIDTH - 1) / TILE_WIDTH; ++k) {
        // Efficiently load tiles into shared memory
        if (k * TILE_WIDTH + tx < colM1 && row < rowM1) {
            Mds[ty][tx] = Mat1[row * colM1 + k * TILE_WIDTH + tx];
        } else {
            Mds[ty][tx] = 0.0f; // Use 0.0f to denote float constant
        }

        if (k * TILE_WIDTH + ty < colM1 && col < colM2) {
            Nds[ty][tx] = Mat2[(k * TILE_WIDTH + ty) * colM2 + col];
        } else {
            Nds[ty][tx] = 0.0f; // Use 0.0f to denote float constant
        }

        __syncthreads();

        // Compute partial product for the current tile
        for (int n = 0; n < TILE_WIDTH; ++n) {
            Pvalue += Mds[ty][n] * Nds[n][tx];
        }

        __syncthreads();
    }

    // Write the block/sub-matrix result to the global memory
    if (row < rowM1 && col < colM2) {
        Mat3[row * colM2 + col] = Pvalue;
    }
}