#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sobelFilterShared2(unsigned char *data, unsigned char *result, int width, int height) {
    // Data cache: threadIdx.x , threadIdx.y
    int ty = threadIdx.y;
    int tx = threadIdx.x;

    // Shared memory represented here by 1D array
    const int n = Mask_size / 2;
    __shared__ int s_data[BLOCKSIZE * (BLOCKSIZE + Mask_size * 2)];

    // Global memory address of the current thread in the whole grid
    const int pos = tx + blockIdx.x * blockDim.x + ty * width + blockIdx.y * blockDim.y * width;

    // Load cache (32x32 shared memory, 16x16 threads blocks)
    int y0 = ty + blockIdx.y * blockDim.y;
    int y = y0 - n;

    if (y >= 0 && y0 < height) {
        // Load upper half into shared memory
        s_data[tx + ty * BLOCKSIZE] = data[pos - (width * n)];
        // Load lower half into shared memory
        s_data[tx + (ty + n) * BLOCKSIZE] = data[pos + (width * n)];
    } else {
        s_data[tx + ty * BLOCKSIZE] = 0;
        s_data[tx + (ty + n) * BLOCKSIZE] = 0;
    }

    __syncthreads();

    // Convolution using optimized memory accesses
    int sum = 0;
    for (int i = 0; i <= n * 2; i++)
        sum += s_data[tx + (ty + i) * BLOCKSIZE] * Global_Mask[i];

    result[pos] = sum;
}