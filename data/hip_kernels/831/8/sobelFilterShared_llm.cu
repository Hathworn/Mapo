#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCKSIZE 16
#define Mask_size 3

__global__ void sobelFilterShared(unsigned char *data, unsigned char *result, int width, int height) {
    // Data cache: threadIdx.x , threadIdx.y
    const int n = Mask_size / 2;
    __shared__ int s_data[BLOCKSIZE + Mask_size * 2][BLOCKSIZE + Mask_size * 2];

    // global mem address of the current thread in the whole grid
    const int x0 = threadIdx.x + blockIdx.x * blockDim.x;
    const int y0 = threadIdx.y + blockIdx.y * blockDim.y;
    const int pos = x0 + y0 * width;

    // Load data from global memory into shared memory
    for (int dy = -n; dy <= n; ++dy) {
        for (int dx = -n; dx <= n; ++dx) {
            int xi = x0 + dx;
            int yi = y0 + dy;
            int shared_x = threadIdx.x + dx + n;
            int shared_y = threadIdx.y + dy + n;
            if (xi < 0 || xi >= width || yi < 0 || yi >= height) {
                // Boundary check
                s_data[shared_y][shared_x] = 0;
            } else {
                // Load valid data
                s_data[shared_y][shared_x] = data[xi + yi * width];
            }
        }
    }

    __syncthreads();

    // Convolution
    int sum = 0;
    int x = n + threadIdx.x;
    int y = n + threadIdx.y;
    for (int i = -n; i <= n; i++) {
        for (int j = -n; j <= n; j++) {
            sum += s_data[y + i][x + j] * Global_Mask[n + i] * Global_Mask[n + j];
        }
    }

    result[pos] = sum;
}