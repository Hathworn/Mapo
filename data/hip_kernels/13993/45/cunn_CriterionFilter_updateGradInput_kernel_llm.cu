#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_CriterionFilter_updateGradInput_kernel( float *gradInput, float *target, float *ignored_label, int batch_size, int n_classes, int map_nelem, int blocks_per_sample)
{
    // Calculate sample and element indices using grid and block dimensions
    int sample = blockIdx.x / blocks_per_sample;
    int elementIndex = (blockIdx.x % blocks_per_sample) * blockDim.x + threadIdx.x;
    int toffset = sample * map_nelem;
    int ioffset = sample * map_nelem * n_classes;
    int ignored_label_num = (int)(ignored_label[0]);

    // Loop through the elements for the current thread with stride optimization
    while (elementIndex < map_nelem) {
        int t = (int)target[toffset + elementIndex];
        if (t == ignored_label_num) {
            // Inline loop for efficiency
            for (int j = 0; j < n_classes; j++) {
                gradInput[ioffset + j * map_nelem + elementIndex] = 0;
            }
        }
        // Advance to the next elementIndex handled by this thread
        elementIndex += blockDim.x * blocks_per_sample;
    }
}