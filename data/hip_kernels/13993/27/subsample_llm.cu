#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void subsample(float *input, float *output, float *weight, float *bias, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Calculate block and thread IDs for output image indexing
    int o = blockIdx.x; 
    int k = blockIdx.x % input_n;

    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Select input/output plane and apply bias and weight
    output += o * output_w * output_h;
    input += o * input_w * input_h; 
    float the_weight = weight[k];
    float the_bias = bias[k];

    // Calculate start offsets and steps for threads
    int xx_start = threadIdx.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    
    // Prefetching pointers to improve memory access patterns
    float *ptr_output = output + yy_start * output_w + xx_start;
    float *ptr_input = input + yy_start * dH * input_w + xx_start * dW;

    // Loop through and accumulate sum
    float sum = 0.0f;
    for (int ky = 0; ky < kH; ++ky) {
        for (int kx = 0; kx < kW; ++kx) {
            sum += ptr_input[kx];
        }
        ptr_input += input_w; // Move to the next row of input
    }

    // Calculate and store the result with the bias and weight applied
    *ptr_output = the_weight * sum + the_bias;
}