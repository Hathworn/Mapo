#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void subgradinputAtomic(float *gradInput, float *gradOutput, float *weight, int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW)
{
    // Output size
    int output_w = (input_w - kW) / dW + 1;
    int output_h = (input_h - kH) / dH + 1;

    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int k = blockIdx.x % input_n;

    // Set starting points for threads
    int xx_start = threadIdx.x;
    int xx_step = blockDim.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;

    // Get weight
    float the_weight = weight[k];

    // Compute gradInput
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float z = *ptr_gradOutput * the_weight;

            int base_idx = (yy * dH) * input_w + (xx * dW);
            for (int ky = 0; ky < kH; ky++) {
                for (int kx = 0; kx < kW; kx++) {
                    // Reduce index calculation within innermost loop
                    int index = base_idx + ky * input_w + kx;
                    atomicAdd(&(gradInput[index]), z);
                }
            }
        }
    }
}