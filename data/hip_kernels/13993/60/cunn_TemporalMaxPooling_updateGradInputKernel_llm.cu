#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(
    float *gradInput, float *gradOutput, float *indices, 
    int input_w, int input_n, int output_w, int kW, int dW) 
{
    // Calculate global thread index
    int timeIndex = threadIdx.x + blockIdx.y * blockDim.x;

    // Ensure the thread index is within bounds
    if (timeIndex < output_w) {
        // Calculate base pointers for each array
        float *gradInput_data = gradInput + blockIdx.x * input_w * input_n 
                                + timeIndex * input_n * dW;
        float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n 
                                 + timeIndex * input_n;
        float *indices_data = indices + blockIdx.x * output_w * input_n 
                              + timeIndex * input_n;
        
        // Iterate over input features and perform updates
        for (int feat = 0; feat < input_n; ++feat) {
            // Use atomicAdd for safe concurrent updates
            atomicAdd(&gradInput_data[(int)indices_data[feat] * input_n + feat], 
                      gradOutput_data[feat]);
        }
    }
}