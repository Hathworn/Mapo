#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void atomicadaptivemaxgradinput( float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w )
{
    // Compute offsets based on thread/block ID
    int o = blockIdx.x;
    int i = o;
    int xx_start = threadIdx.x;
    int yy_start = blockDim.y * blockIdx.y + threadIdx.y;
    int xx_step = blockDim.x;
    int yy_step = blockDim.y * gridDim.y;

    // Select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // Compute gradInput
    for (int yy = yy_start; yy < output_h; yy += yy_step) {
        int y_start = (int)floorf(float(yy) / output_h * input_h);

        for (int xx = xx_start; xx < output_w; xx += xx_step) {
            int x_start = (int)floorf(float(xx) / output_w * input_w);

            float *ptr_gradInput = gradInput + y_start * input_w + x_start;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx;
            float *ptr_ind_x = indices_x + yy * output_w + xx;
            float *ptr_ind_y = indices_y + yy * output_w + xx;
            float z = *ptr_gradOutput;

            int argmax_x = (*ptr_ind_x) - 1;
            int argmax_y = (*ptr_ind_y) - 1;

            // Use atomic add since different threads could update the same variable
            atomicAdd(&(ptr_gradInput[argmax_x + argmax_y * input_w]), z);
        }
    }
}