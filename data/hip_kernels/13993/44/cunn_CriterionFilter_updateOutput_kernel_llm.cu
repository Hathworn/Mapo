#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cunn_CriterionFilter_updateOutput_kernel(float *target, float *ignored_label, int bound, int batch_size, int map_nelem, int blocks_per_sample)
{
    int sample = blockIdx.x / blocks_per_sample;
    int thread_id = threadIdx.x + (blockIdx.x % blocks_per_sample) * blockDim.x;
    int step = blockDim.x * blocks_per_sample;
    int toffset = sample * map_nelem;
    int ignored_label_num = static_cast<int>(ignored_label[0]);

    // Use thread_id directly for more efficient loop iteration
    for (int i = thread_id; i < map_nelem; i += step) {
        if (target[toffset + i] == ignored_label_num) {
            target[toffset + i] = static_cast<float>(bound + 1);
        }
    }
}