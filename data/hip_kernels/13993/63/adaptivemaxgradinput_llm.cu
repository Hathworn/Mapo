#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adaptivemaxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y, int input_n, int input_h, int input_w, int output_h, int output_w)
{
    // compute block index
    int o = blockIdx.x;
    int i = o;

    // compute thread index for y and x
    int xx = threadIdx.x;
    int yy = blockDim.y * blockIdx.y + threadIdx.y;

    // select input/output plane
    gradOutput += o * output_w * output_h;
    gradInput += i * input_w * input_h;
    indices_x += o * output_w * output_h;
    indices_y += o * output_w * output_h;

    // loop over the output height
    while (yy < output_h) {
        int y_start = (int)floorf(float(yy) / output_h * input_h);

        // loop over the output width
        int xx_iter = xx;
        while (xx_iter < output_w) {
            int x_start = (int)floorf(float(xx_iter) / output_w * input_w);

            float *ptr_gradInput = gradInput + y_start * input_w + x_start;
            float *ptr_gradOutput = gradOutput + yy * output_w + xx_iter;
            float *ptr_ind_x = indices_x + yy * output_w + xx_iter;
            float *ptr_ind_y = indices_y + yy * output_w + xx_iter;
            float z = *ptr_gradOutput;

            int argmax_x = (*ptr_ind_x) - 1;
            int argmax_y = (*ptr_ind_y) - 1;

            atomicAdd(&ptr_gradInput[argmax_x + argmax_y * input_w], z); // Use atomicAdd for potential race conditions

            xx_iter += blockDim.x;
        }
        yy += blockDim.y * gridDim.y;
    }
}