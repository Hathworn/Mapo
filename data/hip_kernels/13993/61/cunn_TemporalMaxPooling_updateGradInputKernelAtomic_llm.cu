#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
    // Calculate global index considering both block and thread indices
    int globalIndex = threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS;

    if (globalIndex < output_w) {
        // Calculate the base pointers for the current block
        float *gradInput_data = gradInput + blockIdx.x * input_w * input_n + globalIndex * input_n * dW;
        float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + globalIndex * input_n;
        float *indices_data = indices + blockIdx.x * output_w * input_n + globalIndex * input_n;

        // Use registers to reduce repeated indexing inside the loop
        for (int feat = 0; feat < input_n; ++feat) {
            int gradInputIdx = static_cast<int>(indices_data[feat]) * input_n + feat;
            atomicAdd(&gradInput_data[gradInputIdx], gradOutput_data[feat]);
        }
    }
}