#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define IDX(X, Y, Z, W, D1, D2, D3) (((X) * (D1) + (Y)) * (D2) + (Z)) * (D3) + (W)

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor)
{
    int x, y, z, w;
    w = ii % d3;
    ii /= d3;
    z = ii % d2;
    ii /= d2;
    y = ii % d1;
    ii /= d1;
    x = ii;
    w /= scale_factor;
    z /= scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return IDX(x, y, z, w, d1, d2, d3);
}

__global__ void upscale(float *input, float *output, long no_elements, int scale_factor, int d1, int d2, int d3)
{
    // Calculate global thread index
    long ii = threadIdx.x + blockDim.x * (blockIdx.x + gridDim.x * blockIdx.y);
    ii += threadIdx.y * blockDim.y * (blockDim.x * gridDim.x);

    // Check if the index is within the bounds
    if (ii >= no_elements) return;

    // Translate index and copy value
    int ipidx = translate_idx(ii, d1, d2, d3, scale_factor);
    output[ii] = input[ipidx];
}