#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Perform division using device function to avoid race condition
    atomicExch(accuracy, *accuracy / N);
}