#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void softmax_gradient_kernel(const int dim, const float* Y, const float* dY, float* dX) {
    Y += blockIdx.x * dim;
    dY += blockIdx.x * dim;
    dX += blockIdx.x * dim;
    const int idx = threadIdx.x;
    __shared__ float reduction_buffer[SOFTMAX_NUM_THREADS];

    float sum = 0.0f;

    // First level reduction using warp-synchronous method for efficiency.
    for (int i = idx; i < dim; i += blockDim.x) {
        sum += dY[i] * Y[i];
    }
    reduction_buffer[idx] = sum;

    // Warp-level reduction
    for (int offset = warpSize / 2; offset > 0; offset /= 2) {
        sum += __shfl_down(sum, offset);
    }

    if ((idx % warpSize) == 0) {
        atomicAdd(&reduction_buffer[0], sum);  // Use atomic operation for inter-warp summation.
    }

    __syncthreads();

    // Compute gradient after reduction is finalized.
    float reduction_sum = reduction_buffer[0];
    for (int i = idx; i < dim; i += blockDim.x) {
        dX[i] = Y[i] * (dY[i] - reduction_sum);
    }
}