#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernelB(float* r, float* x, float* y, float* z, int size)
{
    // Use global thread index for improved access pattern
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop with stride to cover large arrays
    for (int i = idx; i < size; i += stride) {
        r[i] = x[i] * y[i] + z[i];
    }
}