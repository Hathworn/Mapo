#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GetOutLod(const size_t* num_erased, const size_t* in_lod, const size_t lod_len, size_t* out_lod0) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the index is within bounds before computations
    if (index < lod_len) {
        size_t lod_value = in_lod[index]; // Cache in_lod value to reduce memory access
        out_lod0[index] = lod_value - num_erased[lod_value];
    }
}