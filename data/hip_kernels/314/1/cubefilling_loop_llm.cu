#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cubefilling_loop(const float* __restrict__ image, float * __restrict__ dev_cube_wi, float * __restrict__ dev_cube_w, const dim3 image_size, int scale_xy, int scale_eps, dim3 dimensions_down)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i < dimensions_down.x && j < dimensions_down.y) {
        size_t cube_idx_1 = i + dimensions_down.x * j;
        
        // Precompute array index offsets for efficiency
        int i_offset = scale_xy * i;
        int j_offset = scale_xy * j;
        
        #pragma unroll
        for (int ii = 0; ii < scale_xy; ii++) {
            int i_idx = i_offset + ii;
            if (i_idx < image_size.x) {
                #pragma unroll
                for (int jj = 0; jj < scale_xy; jj++) {
                    int j_idx = j_offset + jj;
                    if (j_idx < image_size.y) {
                        
                        // Utilize efficient floored division
                        float k = image[i_idx + image_size.x * j_idx];
                        size_t z_index = __float2int_rd(k / scale_eps);
                        
                        if (z_index < dimensions_down.z) {
                            size_t cube_idx_2 = cube_idx_1 + dimensions_down.x * dimensions_down.y * z_index;
                            // Use atomic operations to avoid data races
                            atomicAdd(&dev_cube_wi[cube_idx_2], k);
                            atomicAdd(&dev_cube_w[cube_idx_2], 1.0f);
                        }
                    }
                }
            }
        }
    }
}