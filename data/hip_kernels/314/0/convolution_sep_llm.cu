#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

/*Performs separable convolution on 3d cube*/

__global__ void convolution_sep(float *output, const float *input, const float *kernel, const int kernel_size, const dim3 imsize, int dir)
{
    size_t ix, iy, iz;
    if (dir == X_DIR)
    {
        ix = blockDim.x*blockIdx.x + threadIdx.x;
        iy = blockDim.y*blockIdx.y + threadIdx.y;
        iz = blockIdx.z;
    }
    else if (dir == Y_DIR)
    {
        iy = blockDim.x*blockIdx.x + threadIdx.x;
        ix = blockDim.y*blockIdx.y + threadIdx.y;
        iz = blockIdx.z;
    }
    else if (dir == EPS_DIR)
    {
        iz = blockDim.x*blockIdx.x + threadIdx.x;
        ix = blockDim.y*blockIdx.y + threadIdx.y;
        iy = blockIdx.z;
    }

    const bool valid = ix < imsize.x && iy < imsize.y && iz < imsize.z;
    const size_t cube_idx = ix + iy*imsize.x + iz*imsize.x*imsize.y;

    const size_t radius_size = kernel_size / 2;

    extern __shared__ float s_image[]; // Shared memory for image block
    const size_t s_dim_x = blockDim.x + 2 * radius_size;
    const size_t s_ix = radius_size + threadIdx.x;
    const size_t s_iy = threadIdx.y;

    float result = 0.0;

    // Load input into shared memory for valid region, zero-padding at bounds
    if (threadIdx.x < radius_size) s_image[s_ix - radius_size + s_iy * s_dim_x] = 0.0f;
    if (threadIdx.x >= blockDim.x - radius_size) s_image[s_ix + radius_size + s_iy * s_dim_x] = 0.0f;

    s_image[s_ix + s_iy * s_dim_x] = (valid) ? input[cube_idx] : 0.0f;

    __syncthreads();

    // Perform convolution computation
    #pragma unroll
    for (int i = 0; i < kernel_size; i++)
    {
        result += kernel[i] * s_image[s_ix - i + radius_size + s_iy * s_dim_x];
    }

    // Write result back to global memory if within valid region
    if (valid)
    {
        output[cube_idx] = result;
    }
}