#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matmul(const float_t *A, const float_t *B, float_t *C, const float_t alpha, const float_t beta, int n, int d, int k, int max_block_rows) {
    extern __shared__ __align__(sizeof(float_t)) unsigned char my_smem[];
    float_t *shared = reinterpret_cast<float_t *>(my_smem);

    float_t *s_A = shared;
    float_t *s_B = shared + max_block_rows * d;

    // Load B matrix into shared memory
    for (int idx = threadIdx.x; idx < d * k; idx += blockDim.x) {
        s_B[idx] = B[idx];
    }

    size_t block_start_row_index = blockIdx.x * max_block_rows;
    size_t block_rows = max_block_rows;

    if (blockIdx.x == gridDim.x - 1 && n % max_block_rows != 0) {
        block_rows = n % max_block_rows;
    }

    // Load part of A matrix into shared memory
    for (size_t idx = threadIdx.x; idx < d * block_rows; idx += blockDim.x) {
        s_A[idx] = alpha * A[d * block_start_row_index + idx];
    }

    __syncthreads();

    // Ensure thread responsibility is within the problem bounds
    int col_c = threadIdx.x % k;
    size_t abs_row_c = block_start_row_index + threadIdx.x / k;
    if (abs_row_c >= n || threadIdx.x >= block_rows * k) {
        return;
    }

    float_t elem_c = 0;
    int row_c = threadIdx.x / k;

    // Compute matrix multiplication result
    for (size_t i = 0; i < d; i++) {
        elem_c += s_B[d * col_c + i] * s_A[d * row_c + i];
    }

    // Write result to C matrix with modification of existing values
    atomicAdd(&C[col_c * n + abs_row_c], beta * C[col_c * n + abs_row_c] + elem_c);
}