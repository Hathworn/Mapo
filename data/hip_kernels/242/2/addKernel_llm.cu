#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(float *c, float *a, float *b, int size)
{
    // Compute global thread index for 2D grid of 2D blocks
    int i = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    
    // Use stride loop for efficient global memory access
    int stride = gridDim.x * blockDim.x * blockDim.y;

    while(i < size)
    {
        c[i] = a[i] + b[i];
        i += stride;
    }
}