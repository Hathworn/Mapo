#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_backward(float *ingrad, float *outgrad, float *indata, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for faster access
    extern __shared__ float data[];
    data[threadIdx.x] = indata[tid];
    __syncthreads();

    if (tid < N) {
        // Simplified conditional assignment
        ingrad[tid] = (data[threadIdx.x] > 0) * outgrad[tid];
    }
}