#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

#define Width 1920
#define Height 2520
#define iterations 100

__global__ void convolution_kernel(unsigned char* A, unsigned char* B)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    int x = i - 2 * blockIdx.x - 1;
    int y = j - 2 * blockIdx.y - 1;

    __shared__ unsigned char As[34][34]; // Increase shared memory size for halo

    // Load to shared memory with boundary checks
    if (x < 0) x = 0;
    else if (x >= Width) x = Width - 1;
    if (y < 0) y = 0;
    else if (y >= Height) y = Height - 1;
    As[threadIdx.x + 1][threadIdx.y + 1] = A[Width * y + x];

    // Load halo elements
    if (threadIdx.x == 0 && x > 0) {
        As[0][threadIdx.y + 1] = A[Width * y + (x - 1)];
    }
    if (threadIdx.x == blockDim.x - 1 && x < Width - 1) {
        As[33][threadIdx.y + 1] = A[Width * y + (x + 1)];
    }
    if (threadIdx.y == 0 && y > 0) {
        As[threadIdx.x + 1][0] = A[Width * (y - 1) + x];
    }
    if (threadIdx.y == blockDim.y - 1 && y < Height - 1) {
        As[threadIdx.x + 1][33] = A[Width * (y + 1) + x];
    }

    // Load corners
    if (threadIdx.x == 0 && threadIdx.y == 0 && x > 0 && y > 0) {
        As[0][0] = A[Width * (y - 1) + (x - 1)];
    }
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0 && x < Width - 1 && y > 0) {
        As[33][0] = A[Width * (y - 1) + (x + 1)];
    }
    if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1 && x > 0 && y < Height - 1) {
        As[0][33] = A[Width * (y + 1) + (x - 1)];
    }
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1 && x < Width - 1 && y < Height - 1) {
        As[33][33] = A[Width * (y + 1) + (x + 1)];
    }

    __syncthreads();

    // Computations
    if (x > 0 && x < Width - 1 && y > 0 && y < Height - 1) {
        B[Width * y + x] = (As[threadIdx.x][threadIdx.y] +
                            As[threadIdx.x + 1][threadIdx.y - 1] * 2 +
                            As[threadIdx.x + 2][threadIdx.y] +
                            As[threadIdx.x][threadIdx.y + 1] * 2 +
                            As[threadIdx.x + 1][threadIdx.y + 1] * 4 +
                            As[threadIdx.x + 2][threadIdx.y + 1] * 2 +
                            As[threadIdx.x][threadIdx.y + 2] +
                            As[threadIdx.x + 1][threadIdx.y + 2] * 2 +
                            As[threadIdx.x + 2][threadIdx.y + 2]) / 16;
    }
}