#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    // Optimize data indexing and loop order to improve memory access patterns
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;
    for (uint i = idx; i < tgtWidth * tgtHeight; i += stride) {
        // Compute target coordinates
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Efficiently map target coordinates to source coordinates
        tgt[i] = src[(y % srcHeight) * srcWidth + (x % srcWidth)];
    }
}