#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Compute thread id using grid and block indices
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    // Initialize CURAND state for each thread
    hiprand_init(seed, tidx, 0, &state[tidx]);
}