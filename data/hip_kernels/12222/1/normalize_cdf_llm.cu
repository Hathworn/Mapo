#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Normalize CDF kernel using shared memory for optimization
__global__ void normalize_cdf(unsigned int* d_input_cdf, float* d_output_cdf, int n)
{
    __shared__ float shared_normalization_constant;
    if (threadIdx.x == 0) {
        shared_normalization_constant = 1.f / d_input_cdf[n - 1];
    }
    __syncthreads(); // Ensure all threads have the latest normalization constant

    int global_index_1d = blockIdx.x * blockDim.x + threadIdx.x;

    if (global_index_1d < n) {
        unsigned int input_value = d_input_cdf[global_index_1d];
        float output_value = input_value * shared_normalization_constant;

        d_output_cdf[global_index_1d] = output_value;
    }
}