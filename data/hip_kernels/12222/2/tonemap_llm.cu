#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void tonemap( float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new, float  min_log_Y, float  max_log_Y, float  log_Y_range, int    num_bins, int    num_pixels_y, int    num_pixels_x )
{
    int nx = num_pixels_x;
    int ny = num_pixels_y;

    // Calculate 1D index and check bounds for early return
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    if (x >= nx || y >= ny) return;
    
    int image_index_1d = (nx * y) + x;

    float value_x = d_x[image_index_1d];
    float value_y = d_y[image_index_1d];
    float value_log_Y = d_log_Y[image_index_1d];

    // Direct calculation of bin_index with fminf to avoid extra comparison
    int bin_index = fminf(num_bins - 1, int((num_bins * (value_log_Y - min_log_Y)) / log_Y_range));
    float Y_new = d_cdf_norm[bin_index];

    float Y_over_y = Y_new / value_y;  // Pre-compute reused ratio 
    float X_new = value_x * Y_over_y;
    float Z_new = (1.0f - value_x - value_y) * Y_over_y;

    // Streamlined computation to remove redundancy
    d_r_new[image_index_1d] = fmaf(X_new, 3.2406f, fmaf(Y_new, -1.5372f, Z_new * -0.4986f));
    d_g_new[image_index_1d] = fmaf(X_new, -0.9689f, fmaf(Y_new, 1.8758f, Z_new * 0.0415f));
    d_b_new[image_index_1d] = fmaf(X_new, 0.0557f, fmaf(Y_new, -0.2040f, Z_new * 1.0570f));
}