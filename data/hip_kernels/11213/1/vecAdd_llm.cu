#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecAdd(float* A, float* B, float* C) {
    // Use blockDim.x to support arbitrary numbers of threads per block
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Initialize C using A and B directly; Avoid redundant operations
    C[i] = 0; 
}