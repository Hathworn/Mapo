#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uchar4tofloat4(uchar4 *inputImage, float4 *outputImage, int width, int height) {
    // Calculate global thread indices.
    int offsetX = blockIdx.x * blockDim.x + threadIdx.x;
    int offsetY = blockIdx.y * blockDim.y + threadIdx.y;

    // Compute linear index based on 2D thread index.
    int offset = offsetY * width + offsetX;

    // Check boundary conditions.
    if (offsetX < width && offsetY < height) {
        uchar4 pixel = inputImage[offset];

        // Directly convert to float4 and store.
        outputImage[offset] = make_float4(pixel.x, pixel.y, pixel.z, pixel.w);
    }
}