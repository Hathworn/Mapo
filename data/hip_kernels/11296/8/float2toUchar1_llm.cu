#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void float2toUchar1(float2 *inputImage, uchar1 *outputImage, int width, int height, int index) {
    // Optimize thread offset computation for 1D grid
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    if (col < width && row < height) {  // Ensure within image boundaries
        int offset = row * width + col;
        float2 pixelf = inputImage[offset];
        float pixelfIndexed = (index == 0) ? pixelf.x : pixelf.y;
        uchar1 pixel;
        pixel.x = (unsigned char)pixelfIndexed;
        outputImage[offset] = pixel;
    }
}