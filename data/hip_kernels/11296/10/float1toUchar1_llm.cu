#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void float1toUchar1(float1 *inputImage, uchar1 *outputImage, int width, int height) {
    // Calculate the unique global thread ID
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < width && y < height) { // Ensure we are within image bounds
        int offset = y * width + x; // Calculate linear offset
        float1 pixelf = inputImage[offset];
        uchar1 pixel;
        pixel.x = static_cast<unsigned char>(pixelf.x); // Convert float to uchar
        outputImage[offset] = pixel; // Store the result
    }
}