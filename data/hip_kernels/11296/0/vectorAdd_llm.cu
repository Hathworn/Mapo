#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

extern "C" {
}

/**
* CUDA Kernel Device code
*
* Computes the vector addition of A and B into C. The 3 vectors have the same
* number of elements numElements.
*/

typedef struct {
    float *hA, *hB, *hC;
    float *dA, *dB, *dC;
    int element_count;
    size_t vector_bytes;
    int v_threadsPerBlock;
    int v_blocksPerGrid;
    hipStream_t stream;
} ThreadContext;

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
   
    // Unrolling the loop for better performance
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
    if (i + blockDim.x < numElements) 
    {
        C[i + blockDim.x] = A[i + blockDim.x] + B[i + blockDim.x];
    }
    if (i + 2 * blockDim.x < numElements) 
    {
        C[i + 2 * blockDim.x] = A[i + 2 * blockDim.x] + B[i + 2 * blockDim.x];
    }
    if (i + 3 * blockDim.x < numElements) 
    {
        C[i + 3 * blockDim.x] = A[i + 3 * blockDim.x] + B[i + 3 * blockDim.x];
    }
}