#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void float2toUchar4(float2 *inputImage, uchar4 *outputImage, int width, int height, int index) {
    // Calculate global thread offset for a 2D grid efficiently using 1D index
    int offset = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    
    // Linear offset guard for safety
    if (offset < width * height) {
        float2 pixelf = inputImage[offset];
        float pixelfIndexed = (index == 0) ? pixelf.x : pixelf.y;
        unsigned char pixelVal = (unsigned char) abs(pixelfIndexed);
        
        // Avoid redundant calculations
        uchar4 pixel = make_uchar4(pixelVal, pixelVal, pixelVal, pixelVal);
        outputImage[offset] = pixel;
    }
}