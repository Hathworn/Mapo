#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void float4toUchar4(float4 *inputImage, uchar4 *outputImage, int width, int height) {
    // Calculate unique global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Check boundaries
    if (x < width && y < height) {
        int offset = y * width + x;

        // Load and convert pixel values
        float4 pixelf = inputImage[offset];
        uchar4 pixel;
        pixel.x = (unsigned char) pixelf.x;
        pixel.y = (unsigned char) pixelf.y;
        pixel.z = (unsigned char) pixelf.z;
        pixel.w = (unsigned char) pixelf.w;

        // Store the converted pixel
        outputImage[offset] = pixel;
    }
}