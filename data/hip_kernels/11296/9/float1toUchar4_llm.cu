#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void float1toUchar4(float1 *inputImage, uchar4 *outputImage, int width, int height) {
    // Calculate the global thread offset
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure within image boundaries
    if (x < width && y < height) {
        int offset = y * width + x;
        float1 pixelf = inputImage[offset];
        unsigned char value = static_cast<unsigned char>(pixelf.x);

        // Create uchar4 pixel
        uchar4 pixel;
        pixel.x = value;
        pixel.y = value;
        pixel.z = value;
        pixel.w = value;

        // Assign to output
        outputImage[offset] = pixel;
    }
}