#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bpnn_layerforward_CUDA(float *input_cuda, float *output_hidden_cuda, float *input_hidden_cuda, float *hidden_partial_sum, int in, int hid)
{
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int index = (hid + 1) * HEIGHT * by + (hid + 1) * ty + tx + 1 + (hid + 1);
    int index_in = HEIGHT * by + ty + 1;

    __shared__ float input_node[HEIGHT];
    __shared__ float weight_matrix[HEIGHT][WIDTH];

    if (tx == 0) 
        input_node[ty] = input_cuda[index_in];

    __syncthreads();

    weight_matrix[ty][tx] = input_hidden_cuda[index];

    __syncthreads();

    weight_matrix[ty][tx] *= input_node[ty]; // Multiply in shared memory

    __syncthreads();

    // Unroll reduction loop for efficiency
    for (int stride = HEIGHT / 2; stride > 0; stride >>= 1) {
        if (ty < stride)
            weight_matrix[ty][tx] += weight_matrix[ty + stride][tx];
        __syncthreads();
    }

    if (ty == 0) {
        input_hidden_cuda[index] = weight_matrix[0][tx]; // Store final result per thread
    }

    __syncthreads();

    if (tx == 0) {
        hidden_partial_sum[by * hid + ty] = weight_matrix[ty][0]; // Store row-wise result
    }
}
```
