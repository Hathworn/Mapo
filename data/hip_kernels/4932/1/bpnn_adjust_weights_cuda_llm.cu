#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void bpnn_adjust_weights_cuda(float * delta, int hid, float * ly, int in, float * w, float * oldw )
{
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Calculate indices for weight and layer arrays
    int index =  ( hid + 1 ) * HEIGHT * by + ( hid + 1 ) * ty + tx + 1 + ( hid + 1 );
    int index_y = HEIGHT * by + ty + 1;
    int index_x = tx + 1;

    // Cache delta and ly values in shared memory for reduced global memory accesses
    __shared__ float shared_delta[BLOCK_DIM_X];
    __shared__ float shared_ly[BLOCK_DIM_Y];
    
    if (tx < BLOCK_DIM_X)
        shared_delta[tx] = delta[index_x];
    if (ty < BLOCK_DIM_Y)
        shared_ly[ty] = ly[index_y];
    __syncthreads();
    
    float delta_val = shared_delta[tx];
    float ly_val = shared_ly[ty];
    
    // Update weights and momentum
    float weight_update = ETA * delta_val * ly_val + MOMENTUM * oldw[index];
    w[index] += weight_update;
    oldw[index] = weight_update;

    __syncthreads();

    // Handle edge case for ty == 0 and by == 0
    if (ty == 0 && by == 0)
    {
        float bias_update = ETA * delta_val + MOMENTUM * oldw[index_x];
        w[index_x] += bias_update;
        oldw[index_x] = bias_update;
    }
}