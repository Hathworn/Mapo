#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void bpnn_adjust_weights_cuda(float * delta, int hid, float * ly, int in, float * w, float * oldw)
{
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Calculate indices for accessing weights and inputs
    int index = (hid + 1) * HEIGHT * by + (hid + 1) * ty + tx + 1 + (hid + 1);
    int index_y = HEIGHT * by + ty + 1;
    int index_x = tx + 1;
    
    // Use shared memory for storing the results of updated values temporarily
    __shared__ float shared_w[<relevant size>];
    __shared__ float shared_oldw[<relevant size>];

    // Optimize by using register for commonly accessed eta and momentum calculations
    float eta_delta_ly = ETA * delta[index_x] * ly[index_y];
    float momentum_oldw = MOMENTUM * oldw[index];
    
    // Update weights and old weights efficiently
    shared_w[threadIdx.x] = eta_delta_ly + momentum_oldw;
    w[index] += shared_w[threadIdx.x];
    shared_oldw[threadIdx.x] = shared_w[threadIdx.x];
    oldw[index] = shared_oldw[threadIdx.x];

    __syncthreads();

    // Update global weights in a single branch, removing the redundant calculation
    if (ty == 0 && by == 0) {
        eta_delta_ly = ETA * delta[index_x];
        momentum_oldw = MOMENTUM * oldw[index_x];
        
        shared_w[threadIdx.x] = eta_delta_ly + momentum_oldw;
        w[index_x] += shared_w[threadIdx.x];
        shared_oldw[threadIdx.x] = shared_w[threadIdx.x];
        oldw[index_x] = shared_oldw[threadIdx.x];
    }
}