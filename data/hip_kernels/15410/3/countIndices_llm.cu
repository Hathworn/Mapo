#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void countIndices(int *indices, unsigned int *histo, int size) 
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    int min = blockIdx.x * blockDim.x;
    int max = (blockIdx.x + 1) * blockDim.x;

    extern __shared__ unsigned int tmp[];

    tmp[threadIdx.x] = 0;
    __syncthreads();

    // Load balancing by distributing work more evenly among threads
    for (int i = id; i < size; i += gridDim.x * blockDim.x) 
    {
        if (min <= indices[i] && indices[i] < max) 
        {
            atomicAdd(&tmp[indices[i] - min], 1);
        }
    }

    __syncthreads();

    // Write the result from shared memory to the global histogram
    if (threadIdx.x < blockDim.x) 
    {
        histo[id] = tmp[threadIdx.x];
    }
}