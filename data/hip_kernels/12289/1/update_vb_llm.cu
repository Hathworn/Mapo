#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void update_vb(float *d_verts_ptr, int vertex_count, float timeElapsed)
{
    // Calculate the threadId and check the range within vertex_count
    unsigned long long int threadId = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned long long int idx = threadId * 4;

    // Ensure the access is within bounds
    if (threadId < vertex_count)
    {
        // Direct computation and updating to improve memory coalescing
        d_verts_ptr[idx + 0] *= timeElapsed;
        d_verts_ptr[idx + 1] *= timeElapsed;
        d_verts_ptr[idx + 2] *= timeElapsed;
    }
}