#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate flat index
    if (i < n) { // Ensure within bounds before accessing
        binary[i] = (x[i] >= 0) ? 1.0f : -1.0f; // Direct assignment with constant values
    }
}