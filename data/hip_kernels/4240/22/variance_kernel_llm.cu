#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1.F/(batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Declare shared memory for variance computation
    __shared__ float shared_variance[1024];  // Adjust size based on maximum allowable shared memory

    shared_variance[threadIdx.x] = 0;

    // Parallelize inner loops
    for(int j = 0; j < batch; ++j) {
        for(int k = threadIdx.x; k < spatial; k += blockDim.x) {
            int index = j*filters*spatial + i*spatial + k;
            shared_variance[threadIdx.x] += powf((x[index] - mean[i]), 2);
        }
    }
    
    // Reduce shared variance computations
    __syncthreads();
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            shared_variance[threadIdx.x] += shared_variance[threadIdx.x + stride];
        }
        __syncthreads();
    }
    
    // Write final reduced result back to global memory
    if (threadIdx.x == 0) {
        variance[i] = shared_variance[0] * scale;
    }
}