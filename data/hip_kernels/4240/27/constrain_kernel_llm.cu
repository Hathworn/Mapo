#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index with improved readability
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;

    // Check and process only if within bounds
    if(i < N) {
        float val = X[i*INCX];
        // Use temporary variable for result
        X[i*INCX] = fminf(ALPHA, fmaxf(-ALPHA, val));
    }
}