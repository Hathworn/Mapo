#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta) 
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we only access valid array indices
    if (index < N) {
        int f = (index / spatial) % filters;

        // Precompute recurrent values to reduce processing time
        float var_sqrt_inv = 1.F / (sqrtf(variance[f]) + .000001f);
        float spatial_batch_inv = 1.F / (spatial * batch);
        float delta_val = variance_delta[f] * 2. * (x[index] - mean[f]) * spatial_batch_inv;
        
        // Optimize memory usage and computation
        delta[index] = delta[index] * var_sqrt_inv + delta_val + mean_delta[f] * spatial_batch_inv;
    }
}