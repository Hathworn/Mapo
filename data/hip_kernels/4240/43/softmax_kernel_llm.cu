#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(int n, float *input, float temp, float *output)
{
    int i;
    float sum = 0.0f;
    float largest = -INFINITY;

    // Determine the largest element (optimization for numerical stability)
    for (i = 0; i < n; ++i) {
        float val = input[i];
        largest = (val > largest) ? val : largest;
    }

    // Compute exponentials and update the sum (numerical stability improvement)
    for (i = 0; i < n; ++i) {
        float e = expf((input[i] - largest) / temp);  // Simplified exp calculation
        sum += e;
        output[i] = e;
    }

    // Normalize the output
    float invSum = 1.0f / sum;  // Compute reciprocal once for all normalization
    for (i = 0; i < n; ++i) {
        output[i] *= invSum;
    }
}

__global__ void softmax_kernel(int n, int offset, int batch, float *input, float temp, float *output)
{
    int b = blockIdx.x * blockDim.x + threadIdx.x;
    if (b < batch) {
        softmax_device(n, input + b * offset, temp, output + b * offset);
    }
}