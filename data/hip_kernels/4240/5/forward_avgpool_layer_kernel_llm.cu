#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n * c) return;  // Adjusted boundary to n * c

    int k = id % c;
    int b = id / c;

    int out_index = k + c * b;
    float sum = 0.0f;
    for (int i = 0; i < w * h; ++i) {  // Loop directly with w * h
        int in_index = i + w * h * (k + b * c);
        sum += input[in_index];
    }
    output[out_index] = sum / (w * h);  // Compute average directly
}