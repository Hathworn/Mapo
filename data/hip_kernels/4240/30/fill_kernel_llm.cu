#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride to allow every thread to process multiple elements if necessary
    int stride = blockDim.x * gridDim.x;
    for(; i < N; i += stride)
    {
        X[i * INCX] = ALPHA;
    }
}