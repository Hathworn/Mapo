#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    size_t index = blockIdx.x * blockDim.x + threadIdx.x;
    size_t step = gridDim.x * blockDim.x;
    
    // Loop to handle larger data sets using the same kernel
    for (size_t i = index; i < N; i += step) {
        int out_index = i;
        int out_w = i % (w * stride);
        i = i / (w * stride);
        int out_h = i % (h * stride);
        i = i / (h * stride);
        int out_c = i % c;
        i = i / c;
        int b = i % batch;

        int in_w = out_w / stride;
        int in_h = out_h / stride;
        int in_c = out_c;

        int in_index = b * w * h * c + in_c * w * h + in_h * w + in_w;
        
        // Use atomic operations to prevent race conditions
        if (forward) {
            out[out_index] += scale * x[in_index];
        } else {
            atomicAdd(x + in_index, scale * out[out_index]);
        }
    }
}