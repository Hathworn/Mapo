#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int c = in_c;
    
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= n) return;
    
    int out_index = id;
    int j = out_index % w;
    out_index /= w;
    int i = out_index % h;
    out_index /= h;
    int k = out_index % c;
    int b = out_index / c;
    
    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    float max = -INFINITY;
    int max_i = -1;

    // Optimize loop bounds for better register use
    for (int l = 0; l < size; ++l) {
        for (int m = 0; m < size; ++m) {
            int cur_h = h_offset + i * stride + l;
            int cur_w = w_offset + j * stride + m;
            
            if (cur_h >= 0 && cur_h < in_h && cur_w >= 0 && cur_w < in_w) { // Directly check validity
                int index = cur_w + in_w * (cur_h + in_h * (k + b * in_c));
                float val = input[index];
                
                if (val > max) {
                    max = val;
                    max_i = index;
                }
            }
        }
    }
    
    output[id] = max;  // Simplified the output indexing by using id
    indexes[id] = max_i;
}