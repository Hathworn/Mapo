#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    float h = hsv.x;
    float s = hsv.y;
    float v = hsv.z;
    float r, g, b;
    float f, p, q, t;

    if (s == 0) {
        // Achromatic case (saturation is zero)
        r = g = b = v;
    } else {
        // Optimized calculation: using modulo operation for h index
        int index = (int)(h) % 6;
        f = h - (int)(h);
        p = v * (1 - s);
        q = v * (1 - s * f);
        t = v * (1 - s * (1 - f));

        // Reduce repeated evaluations using switch-case
        switch (index) {
            case 0: r = v; g = t; b = p; break;
            case 1: r = q; g = v; b = p; break;
            case 2: r = p; g = v; b = t; break;
            case 3: r = p; g = q; b = v; break;
            case 4: r = t; g = p; b = v; break;
            case 5: r = v; g = p; b = q; break;
        }
    }
    // Clamp results to [0, 1]
    r = fminf(fmaxf(r, 0.0f), 1.0f);
    g = fminf(fmaxf(g, 0.0f), 1.0f);
    b = fminf(fmaxf(b, 0.0f), 1.0f);

    return make_float3(r, g, b);
}

__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    float r = rgb.x;
    float g = rgb.y;
    float b = rgb.z;
    float h, s, v;
    
    // Use fmaxf/fminf for max/min calculation
    float max = fmaxf(r, fmaxf(g, b));
    float min = fminf(r, fminf(g, b));
    float delta = max - min;
    v = max;

    if (max == 0) {
        // Achromatic case (max is zero)
        s = 0;
        h = -1; // Undefined hue
    } else {
        s = delta / max;
        
        // Reduce repeated evaluations using conditional assignment
        if (r == max)
            h = (g - b) / delta;
        else if (g == max)
            h = 2 + (b - r) / delta;
        else
            h = 4 + (r - g) / delta;

        if (h < 0) h += 6;
    }

    return make_float3(h, s, v);
}

__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    
    // Simplified calculation of indices
    int x = id % w;
    int y = (id / w) % h;
    id /= h;

    float rshift = rand[0];
    float gshift = rand[1];
    float bshift = rand[2];
    float r0 = rand[8 * id + 0];
    float r1 = rand[8 * id + 1];
    float r2 = rand[8 * id + 2];
    float r3 = rand[8 * id + 3];

    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > 0.5f) ? 1.0f / saturation : saturation;
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > 0.5f) ? 1.0f / exposure : exposure;

    size_t offset = id * h * w * 3;
    image += offset;

    float r = image[x + w * y];
    float g = image[x + w * (y + h)];
    float b = image[x + w * (y + 2 * h)];
    float3 rgb = make_float3(r, g, b);
    
    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }

    // Simplify write-back with reduced redundancy
    image[x + w * y] = rgb.x * scale + translate + (rshift - 0.5f) * shift;
    image[x + w * (y + h)] = rgb.y * scale + translate + (gshift - 0.5f) * shift;
    image[x + w * (y + 2 * h)] = rgb.z * scale + translate + (bshift - 0.5f) * shift;
}