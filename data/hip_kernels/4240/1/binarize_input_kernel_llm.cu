#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}


__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = blockIdx.x * blockDim.x + threadIdx.x; // Simplified block and thread index calculation
    if (s >= size) return;
    
    float mean = 0.0f;
    for (int i = 0; i < n; ++i)
    {
        mean += fabs(input[i * size + s]); // Accumulate absolute values
    }
    mean /= n; // Compute mean
    
    for (int i = 0; i < n; ++i)
    {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean; // Binarize input
    }
}