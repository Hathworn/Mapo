#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int i, b;
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f;

    // Unroll loop for better performance
    for (b = 0; b < batch; ++b) {
        for (i = 0; i < size; i += BLOCK) {
            int index = p + i + size * (filter + n * b);
            if (p + i < size) {
                sum += delta[index] * x_norm[index];
            }
        }
    }

    part[p] = sum;
    __syncthreads();

    // Use reduction to accumulate results
    if (p < 32) {
        part[p] += part[p + 32];
        part[p] += part[p + 16];
        part[p] += part[p + 8];
        part[p] += part[p + 4];
        part[p] += part[p + 2];
        part[p] += part[p + 1];
    }

    __syncthreads();

    // Write the result back to global memory
    if (p == 0) {
        atomicAdd(&scale_updates[filter], part[0]);
    }
}