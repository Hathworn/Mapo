#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#ifdef CUDNN
#pragma comment(lib, "cudnn.lib")
#endif

extern "C" {
}

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation for 1D grid
    if (f >= n) return;

    // Allocate shared memory for partial sums
    __shared__ float partial_sum[1024]; // Adjust size according to blockDim.x if needed
    partial_sum[threadIdx.x] = 0.0f;

    int i = threadIdx.x;
    while (i < size) {
        partial_sum[threadIdx.x] += fabs(weights[f*size + i]);
        i += blockDim.x;
    }
    __syncthreads();

    // Reduce sum in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
        }
        __syncthreads();
    }

    float mean = partial_sum[0] / size; // Only the first thread has the correct mean

    // Reuse mean computation for binarization
    i = threadIdx.x;
    while (i < size) {
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
        i += blockDim.x;
    }
}