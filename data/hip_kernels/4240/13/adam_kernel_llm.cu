#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    // Calculate index within bounds
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    // Precompute terms outside loop to optimize
    float B2_pow_t = powf(B2, t);
    float B1_pow_t = powf(B1, t);
    float one_minus_B2_pow_t = 1.F - B2_pow_t;
    float one_minus_B1_pow_t = 1.F - B1_pow_t;
    float corrected_rate = rate * sqrtf(one_minus_B2_pow_t) / one_minus_B1_pow_t;

    // Update x
    x[index] = x[index] - (corrected_rate * m[index] / (sqrtf(v[index]) + eps));
}