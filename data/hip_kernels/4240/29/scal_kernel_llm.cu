#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Optimize by using a faster calculation for the index.
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    if(i < N) 
    {
        X[i*INCX] *= ALPHA;
    }
}