#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    int batch = blockIdx.z;
    int filter = blockIdx.y;
    int offset = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for biases
    __shared__ float shared_bias;
    if (threadIdx.x == 0) {
        shared_bias = biases[filter];
    }
    __syncthreads();

    if (offset < size) {
        // Prefetch biases and reduce global memory access
        output[(batch*n+filter)*size + offset] += shared_bias;
    }
}