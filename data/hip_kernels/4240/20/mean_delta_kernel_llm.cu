#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Initialize mean_delta to zero using shared memory for faster access
    __shared__ float shared_mean_delta;
    if (threadIdx.x == 0) shared_mean_delta = 0;
    __syncthreads();
    
    // Use atomicAdd to avoid race conditions while updating shared memory
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            atomicAdd(&shared_mean_delta, delta[index]);
        }
    }
    
    // Finalize mean_delta computation after all threads are done updating shared_mean_delta
    if (threadIdx.x == 0) {
        mean_delta[i] = shared_mean_delta * (-1.F/sqrtf(variance[i] + .000001f));
    }
}