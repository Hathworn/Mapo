#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate linear thread index in grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds
    if (i < n)
    {
        // Compute weighted sum with b null-checking
        float b_val = (b != nullptr) ? b[i] : 0.0f;
        c[i] = s[i] * a[i] + (1 - s[i]) * b_val;
    }
}
```
