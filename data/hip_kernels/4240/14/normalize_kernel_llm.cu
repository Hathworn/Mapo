#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Calculate 1D index
    if (index < N) {
        // Precompute filter index
        int f = (index / spatial) % filters;
        
        // Normalize the element
        x[index] = (x[index] - mean[f]) / (sqrtf(variance[f]) + .000001f);
    }
}