#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[BLOCK];  // Fixed size to BLOCK for better consistency
    int id = threadIdx.x;
    int filter = blockIdx.x;
    local[id] = 0.0f;

    int offset = filter * spatial;

    // Accumulate values in shared memory
    for (int j = 0; j < batch; ++j) {
        int base_index = j * spatial * filters + offset;
        for (int i = id; i < spatial; i += threads) {  // Strided access for better utilization
            local[id] += x[base_index + i];
        }
    }

    __syncthreads();

    // Reduce shared memory values to compute mean
    if (id == 0) {
        float sum = 0.0f;
        for (int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        mean[filter] = sum / (spatial * batch);
    }
}