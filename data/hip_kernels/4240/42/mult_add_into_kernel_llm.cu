#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate a linear index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
   
    // Use shared memory for better memory access efficiency
    __shared__ float sharedA[256];
    __shared__ float sharedB[256];

    if(i < n){
        sharedA[threadIdx.x] = a[i];
        sharedB[threadIdx.x] = b[i];
        __syncthreads(); // Ensure all threads have loaded A and B
        
        // Perform operation using shared memory
        c[i] += sharedA[threadIdx.x] * sharedB[threadIdx.x];
    }
}