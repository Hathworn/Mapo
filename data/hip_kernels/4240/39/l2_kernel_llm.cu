#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize thread index calculation

    if(i < n){
        float diff = truth[i] - pred[i];
        error[i] = diff * diff; // Calculate the squared error
        delta[i] = diff; // Store the difference
    }
}