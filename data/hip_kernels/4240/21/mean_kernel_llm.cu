#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Optimize calculation using shared memory
    extern __shared__ float shared_data[];

    float scale = 1.0f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float sum = 0.0f;

    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += x[index];
        }
    }

    shared_data[threadIdx.x] = sum;
    __syncthreads();

    // Use reduction to ensure all results are summed correctly
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (threadIdx.x < s) {
            shared_data[threadIdx.x] += shared_data[threadIdx.x + s];
        }
        __syncthreads();
    }

    // Save the final result to global memory
    if (threadIdx.x == 0) {
        mean[blockIdx.x] = shared_data[0] * scale;
    }
}