#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Efficient boundary check and computation using loop unrolling
    #pragma unroll
    for (; i < N; i += blockDim.x * gridDim.x) {
        Y[i * INCY] = powf(X[i * INCX], ALPHA);
    }
}