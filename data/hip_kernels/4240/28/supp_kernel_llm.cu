#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process if within bounds
    if(i < N) {
        float val = X[i * INCX];
        // Use ALPHA squared to avoid repetitive computation
        if((val * val) < (ALPHA * ALPHA)) X[i * INCX] = 0;
    }
}