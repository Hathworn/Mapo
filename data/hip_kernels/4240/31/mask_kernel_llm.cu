#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel by using blockDim.y and threadIdx.y to improve warp efficiency
__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    int i = ((blockIdx.y * gridDim.x + blockIdx.x) * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
    if (i < n && mask[i] == mask_num) x[i] = mask_num;
}