#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float* X, int OFFX, int INCX, float* Y, int OFFY, int INCY) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;  // Calculate the index using only 1D grid
    if (idx < N) {
        // Simplified access patterns to improve memory coalescing
        Y[idx * INCY + OFFY] = X[idx * INCX + OFFX];  
    }
}