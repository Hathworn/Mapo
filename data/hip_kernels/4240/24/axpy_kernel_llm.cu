#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread ID
    int gridStride = gridDim.x * blockDim.x; // Calculate grid stride

    // Grid-stride loop for improved utilization
    for (; i < N; i += gridStride) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}