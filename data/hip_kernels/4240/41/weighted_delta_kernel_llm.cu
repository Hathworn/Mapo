#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified block index calculation
    if(i < n){
        float dc_i = dc[i];  // Cache dc[i] to avoid repeated access
        float s_i = s[i];    // Cache s[i] to avoid repeated access
        if(da) da[i] += dc_i * s_i;
        db[i] += dc_i * (1 - s_i);
        ds[i] += dc_i * (a[i] - b[i]);  // Combined addition operations
    }
}