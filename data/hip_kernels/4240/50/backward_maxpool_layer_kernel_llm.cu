#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    // Precompute constants outside of loop
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int area = (size - 1) / stride;
    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    // Compute global thread index
    int id = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    // Calculate the coordinates
    int index = id;
    int j = id % in_w; id /= in_w;
    int i = id % in_h; id /= in_h;
    int k = id % in_c; id /= in_c;
    int b = id;

    float d = 0;

    // Unroll loops for optimization
    #pragma unroll
    for(int l = -area; l < area + 1; ++l) {
        #pragma unroll
        for(int m = -area; m < area + 1; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            int out_index = out_w + w * (out_h + h * (k + c * b));
            bool valid = (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h);
            d += (valid && indexes[out_index] == index) ? delta[out_index] : 0;
        }
    }
    
    prev_delta[index] += d;
}