#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (id >= n) return;

    // Compute indices
    int k = id % c;
    int b = id / c;
    int out_index = k + c * b;

    // Cache division result
    float inv_wh = 1.0f / (w * h);

    // Iterate over width*height
    for (int i = 0; i < w * h; ++i) {
        int in_index = i + w * h * (k + b * c);
        // Update delta with pooled average
        in_delta[in_index] += out_delta[out_index] * inv_wh;
    }
}