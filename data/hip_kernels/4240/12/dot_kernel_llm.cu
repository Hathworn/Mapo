#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Calculate unique thread index
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    if (f2 <= f1) return;

    // Initialize accumulators
    float sum = 0.0f;
    float norm1 = 0.0f;
    float norm2 = 0.0f;

    // Loop unrolling potential inside inner loop
    for(int b = 0; b < batch; ++b) {
        for(int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            float out_i1 = output[i1]; // Cache output[i1]
            float out_i2 = output[i2]; // Cache output[i2]
            sum += out_i1 * out_i2;
            norm1 += out_i1 * out_i1;
            norm2 += out_i2 * out_i2;
        }
    }

    // Calculating norms outside loops
    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    sum /= norm;

    for(int b = 0; b < batch; ++b) {
        for(int i = 0; i < size; ++i) {
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            delta[i1] += -scale * sum * output[i2] / norm;
            delta[i2] += -scale * sum * output[i1] / norm;
        }
    }
}