#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation for 1D grid
    if(i < n) {
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);  // Use fast math function for absolute value
        if(abs_val < 1.0f) {  // Use floating point literal
            error[i] = diff * diff;
            delta[i] = diff;
        }
        else {
            error[i] = 2.0f * abs_val - 1.0f;  // Use floating point literals
            delta[i] = copysignf(1.0f, diff);  // Use copysign for branchless sign calculation
        }
    }
}