#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure the loop increments by the grid size
    for( int idx = i; idx < N; idx += blockDim.x * gridDim.x) {
        X[idx * INCX] = ALPHA; // Assign value ALPHA to each relevant index
    }
}