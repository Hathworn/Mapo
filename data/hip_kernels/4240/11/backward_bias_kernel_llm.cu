#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f;

    // Reduce loop iterations by processing multiple elements in a single thread
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Use parallel reduction to accumulate results from shared memory
    for (int stride = BLOCK / 2; stride > 0; stride >>= 1) {
        if (p < stride) {
            part[p] += part[p + stride];
        }
        __syncthreads();
    }

    if (p == 0) {
        bias_updates[filter] += part[0];
    }
}