#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize: Remove blockIdx.y, as it is not needed for 1D grid
    int gridSize = blockDim.x * gridDim.x; // Calculate grid stride for loop to handle larger arrays

    // Use grid-stride loop to allow handling of N elements with fewer threads
    for (; i < N; i += gridSize) 
    {
        Y[i * INCY] *= X[i * INCX];
    }
}