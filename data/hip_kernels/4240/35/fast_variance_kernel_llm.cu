#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    // Loop unrolling
    for(j = 0; j < batch; ++j){
        for(i = id; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i;
            float diff = (i < spatial) ? x[index] - mean[filter] : 0;
            local[id] += diff * diff;
        }
    }
    __syncthreads();

    // Reduce using parallel reduction
    if(id < 32){
        for(i = threads / 2; i > 32; i >>= 1){
            if(id < i){
                local[id] += local[id + i];
            }
            __syncthreads();
        }
        if(id < 32){
            volatile float* vlocal = local;
            vlocal[id] += vlocal[id + 32];
            vlocal[id] += vlocal[id + 16];
            vlocal[id] += vlocal[id + 8];
            vlocal[id] += vlocal[id + 4];
            vlocal[id] += vlocal[id + 2];
            vlocal[id] += vlocal[id + 1];
        }
    }

    if(id == 0){
        variance[filter] = local[0] / (spatial * batch - 1);
    }
}
```
