#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;
    
    // Initialize shared memory with 0
    local[id] = 0;

    for(int j = 0; j < batch; ++j) {
        for(int i = id; i < spatial; i += threads) { // Optimize loop with step size as threads
            int index = j * spatial * filters + filter * spatial + i;
            local[id] += delta[index] * (x[index] - mean[filter]);
        }
    }
    __syncthreads();

    // Accumulate results using only one thread
    if(id == 0){
        variance_delta[filter] = 0;
        for(int i = 0; i < threads; ++i){
            variance_delta[filter] += local[i];
        }
        // Constant expressions computed only once
        float variance_factor = -.5 * powf(variance[filter] + .000001f, (float)(-3./2.));
        variance_delta[filter] *= variance_factor;
    }
}