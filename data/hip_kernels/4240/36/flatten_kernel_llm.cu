#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Use shared memory for faster access and reduce redundant calculations
    extern __shared__ float shared_x[];
    
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= N) return;

    // Pre-calculate repeated expressions
    int layer_stride = layers * spatial;
    int batch_stride = batch * layer_stride;
  
    // Calculate indices
    int in_s = i % spatial;
    int in_c = (i / spatial) % layers;
    int b = i / layer_stride;

    int i1 = b * layer_stride + in_c * spatial + in_s;
    int i2 = b * layer_stride + in_s * layers + in_c;

    // Use local variables to access shared memory
    __shared__ float shared_value1, shared_value2;
    shared_value1 = x[i1];
    shared_value2 = !forward ? x[i2] : 0.0f;

    // Perform the assignment using shared values
    if (forward) out[i2] = shared_value1;
    else out[i1] = shared_value2;
}