#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Calculate the thread's global index
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= groups) return;

    // Initialize sum accumulator to zero
    float local_sum = 0;

    // Use shared memory to optimize memory access within a block
    extern __shared__ float shared_x[];
    for (int k = threadIdx.x; k < n; k += blockDim.x) {
        shared_x[threadIdx.x] = x[k * groups + i];
        __syncthreads(); // Synchronization point

        // Accumulate sum in local variable
        local_sum += shared_x[threadIdx.x];
        __syncthreads(); // Ensure all threads have completed reading and writing
    }

    // Write the result back to global sum array
    sum[i] = local_sum;
}