#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void cuda_Cal_Gfw(float *GfW, float2 *Grf, float2 *Gcf, int nRows, int nCols) {
    unsigned int Tidx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int Tidy = threadIdx.y + blockIdx.y * blockDim.y;

    // Consolidate operations for improved efficiency
    if (Tidx < nCols && Tidy < nRows) {
        unsigned int index = Tidx + Tidy * nCols;
        float2 Grf_temp = Grf[index];
        float2 Gcf_temp = Gcf[index];
        
        // Optimize computation by combining operations
        float GfW_temp = (Grf_temp.x * Grf_temp.x + Grf_temp.y * Grf_temp.y) +
                         (Gcf_temp.x * Gcf_temp.x + Gcf_temp.y * Gcf_temp.y);

        GfW[index] = GfW_temp;
    }
}