#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_Pad_Dict(float *PadD, float *D, int nRows_D, int nCols_D, int nFilts, int nRows, int nCols) {
    // Calculate global thread index
    unsigned int Tidx_D = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int Tidy_D = threadIdx.y + blockIdx.y * blockDim.y;

    int Dim_D = nRows_D * nFilts;

    // Use shared memory to efficiently handle memory accesses
    __shared__ float tile[32][32]; // assuming blockDim.x and blockDim.y <= 32
    if (Tidx_D < nCols_D && Tidy_D < nRows_D) {
        for (int i = Tidy_D, j = Tidy_D; i < Dim_D; i += nRows_D, j += nRows) {
            // Load data into shared memory
            tile[threadIdx.y][threadIdx.x] = D[Tidx_D + i * nCols_D];
            __syncthreads(); // Ensure all data is loaded before accessing

            // Read from shared memory into global memory
            PadD[Tidx_D + j * nCols] = tile[threadIdx.y][threadIdx.x];
            __syncthreads(); // Ensure all data is written before next iteration
        }
    }
}