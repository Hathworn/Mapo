#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_Shrink_CalU_Vector(float *Y, float *U, float *X, float lambda, float *L1Weight, int nRows, int nCols, int nFilts) {
    unsigned int Tidx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int Tidy = threadIdx.y + blockIdx.y * blockDim.y;

    // Ensure threads are within bounds
    if (Tidx >= nCols || Tidy >= nRows) return;

    int nElements = nRows * nCols;
    for (int k = 0; k < nFilts; k++) {
        int index = Tidx + (Tidy + nRows * k) * nCols;

        // Consolidate repeated calculations outside loop
        float factor = 1.0f / nElements;
        float X_temp = X[index] * factor;
        float U_temp = U[index];
        float WLambda = lambda * L1Weight[k];

        float Y_temp = X_temp + U_temp;
        float absxV1 = fabsf(Y_temp) - WLambda;

        // Simplify sign and absolute calculations
        Y_temp = copysignf(fmaxf(absxV1, 0.0f), Y_temp);

        Y[index] = Y_temp;
        U[index] = U_temp + X_temp - Y_temp;
    }
}