#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cuda_Cal_X_minus_U_W(float *Y, float *U, float *X, int *Weight, int nRows, int nCols) {
    unsigned int Tidx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int Tidy = threadIdx.y + blockIdx.y * blockDim.y;
    
    // Calculate global index only once
    unsigned int index = Tidx + Tidy * nCols;

    if (Tidx < nCols && Tidy < nRows) {
        // Directly compute without temporary variables
        float X_temp_U_temp = X[index] / (nRows * nCols) + U[index];

        float Y_temp = (1 - Weight[index]) * X_temp_U_temp;

        // Update global memory
        Y[index] = Y_temp;
        U[index] = X_temp_U_temp - Y_temp;
    }
}