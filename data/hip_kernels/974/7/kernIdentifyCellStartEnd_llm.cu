#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define GLM_FORCE_CUDA

#ifndef imax
#define imax( a, b ) ( ((a) > (b)) ? (a) : (b) )
#endif

#ifndef imin
#define imin( a, b ) ( ((a) < (b)) ? (a) : (b) )
#endif

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

__global__ void kernIdentifyCellStartEnd(int N, int *particleGridIndices, int *gridCellStartIndices, int *gridCellEndIndices) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;

    // Initialize the start and end indices at the first cell
    if (index == 0) {
        gridCellStartIndices[particleGridIndices[0]] = 0;
    }
    if (index == N - 1) {
        gridCellEndIndices[particleGridIndices[index]] = N - 1;
    }

    // Check boundaries between particles to identify cell start and end points
    if (index > 0) {
        int currentCell = particleGridIndices[index];
        int previousCell = particleGridIndices[index - 1];

        if (currentCell != previousCell) {
            gridCellStartIndices[currentCell] = index;  // First particle of a new cell
            gridCellEndIndices[previousCell] = index - 1;  // Last particle of the previous cell
        }
    }
}