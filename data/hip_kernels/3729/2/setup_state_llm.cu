#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel to use a unique thread index for initializing state
__global__ void setup_state(hiprandState* state, unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique thread index
    hiprand_init(seed, idx, 0, &state[idx]);         // Initialize state using unique index
}