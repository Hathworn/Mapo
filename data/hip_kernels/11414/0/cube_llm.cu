#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cube(float *d_out, float *d_in) {
    // Use blockDim.x for better scalability and idx calculation
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure idx doesn't access out-of-bound memory
    if (idx < blockDim.x) {
        float f = d_in[idx];
        d_out[idx] = f * f * f;
    }
}