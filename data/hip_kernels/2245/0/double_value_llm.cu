#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void double_value(double *x, double *y)
{
    // Use blockDim and blockIdx for scalability
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    y[idx] = 2.0 * x[idx]; // Use consistent floating-point literal
}