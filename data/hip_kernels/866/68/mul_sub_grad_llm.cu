#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_sub_grad(float* in1_x, float* in1_d, float* in2_x, float* in2_d, float* out, int in1ScalarCount, int in2ScalarCount) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop to cover all elements
    int stride = gridDim.x * blockDim.x;
    for (int i = tid; i < in1ScalarCount; i += stride) {
        // Calculate index for the in2 array
        int index = i % in2ScalarCount;
        
        // Perform computation and update gradients
        float tmp = out[i];
        atomicAdd(&in1_d[i], tmp * in2_x[index]); // Use atomic add to handle concurrent writes
        in2_d[i] = in1_x[i] * tmp;
    }
}