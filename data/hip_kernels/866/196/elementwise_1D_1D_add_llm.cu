#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel by using shared memory and improved loop condition
__global__ void elementwise_1D_1D_add(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (; tid < size; tid += stride) {
        out[tid] = in1[tid] + in2[tid];
    }
}