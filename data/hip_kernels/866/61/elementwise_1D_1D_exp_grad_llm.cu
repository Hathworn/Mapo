#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_exp_grad(float* in_x, float* in_d, float* out_x, float* out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll the loop for improved hardware utilization
    for (; tid < size; tid += stride * 4) {
        if (tid < size) in_d[tid] += out_d[tid] * out_x[tid];
        if (tid + stride < size) in_d[tid + stride] += out_d[tid + stride] * out_x[tid + stride];
        if (tid + 2 * stride < size) in_d[tid + 2 * stride] += out_d[tid + 2 * stride] * out_x[tid + 2 * stride];
        if (tid + 3 * stride < size) in_d[tid + 3 * stride] += out_d[tid + 3 * stride] * out_x[tid + 3 * stride];
    }
}