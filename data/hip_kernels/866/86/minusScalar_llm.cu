#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void minusScalar(float* in, float* out, float minus, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Unroll the loop to enhance performance
    for (; tid < size; tid += stride * 4) {
        if (tid < size) out[tid] = in[tid] - minus;
        if (tid + stride < size) out[tid + stride] = in[tid + stride] - minus;
        if (tid + 2 * stride < size) out[tid + 2 * stride] = in[tid + 2 * stride] - minus;
        if (tid + 3 * stride < size) out[tid + 3 * stride] = in[tid + 3 * stride] - minus;
    }
}