#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_add(float* in1, float* in2, float* out, int size) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop over the elements using the stride
    for (int i = tid; i < size; i += stride) {
        out[i] = in1[i] + in2[i];
    }
}