#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void mul_sub(float* in1, float* in2, float* out, int in1ScalarCount, int in2ScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use shared memory to reduce global memory access
    __shared__ float sharedIn2[256]; // Assumes blockDim.x <= 256
    if (threadIdx.x < in2ScalarCount) {
        sharedIn2[threadIdx.x] = in2[threadIdx.x];
    }
    __syncthreads();

    for (; tid < in1ScalarCount; tid += stride) {
        out[tid] = in1[tid] * sharedIn2[tid % in2ScalarCount];
    }
}