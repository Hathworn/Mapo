#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_log(float* in, float* out, int size) {
    // Calculate unique global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure no overrun condition by using a single check
    if (tid < size) {
        out[tid] = log(in[tid]);
    }
    
    // Utilize stride for more threads
    int stride = gridDim.x * blockDim.x;
    for (tid += stride; tid < size; tid += stride) {
        out[tid] = log(in[tid]);
    }
}