#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_sub(float* in1, float* in2, float* out, int in1ScalarCount, int in2ScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use loop unrolling to optimize performance
    for (int i = tid; i < in1ScalarCount; i += stride * 4) {
        if (i < in1ScalarCount) {
            out[i] = in1[i] * in2[i % in2ScalarCount];
        }
        if (i + stride < in1ScalarCount) {
            out[i + stride] = in1[i + stride] * in2[(i + stride) % in2ScalarCount];
        }
        if (i + 2 * stride < in1ScalarCount) {
            out[i + 2 * stride] = in1[i + 2 * stride] * in2[(i + 2 * stride) % in2ScalarCount];
        }
        if (i + 3 * stride < in1ScalarCount) {
            out[i + 3 * stride] = in1[i + 3 * stride] * in2[(i + 3 * stride) % in2ScalarCount];
        }
    }
}