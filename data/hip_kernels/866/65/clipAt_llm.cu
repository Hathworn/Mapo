#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clipAt(float* in, float bound, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll loops for more efficient parallel execution
    for (; tid < size; tid += stride) {
        float value = in[tid];
        
        if (value > bound) {
            in[tid] = bound;
        } else if (value < -bound) {
            in[tid] = -bound;
        }
    }
}