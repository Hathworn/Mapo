#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void repeat0(float* in, float* out, int outStride0, int outStride1, int outScalarCount) {
    // Calculate the global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    int linearIndex, outIndex0, outIndex1, outIndex2, inIndex;
    
    // Use shared memory if beneficial (depending on data access patterns - check separately if there's a benefit)
    // __shared__ float tile[blockDim.x];

    for (; tid < outScalarCount; tid += stride) {
        linearIndex = tid;

        // Optimize division and modulo using intrinsic functions if available
        outIndex0 = __div_trunc(linearIndex, outStride0);
        linearIndex -= outIndex0 * outStride0;
        
        outIndex1 = __div_trunc(linearIndex, outStride1);
        outIndex2 = linearIndex - outIndex1 * outStride1;
        
        inIndex = outIndex2 + (outIndex0 + outIndex1) * outStride1;
        
        // Read input and write output; confirm whether coalescing and access patterns are efficient
        out[tid] = in[inIndex];
    }
}