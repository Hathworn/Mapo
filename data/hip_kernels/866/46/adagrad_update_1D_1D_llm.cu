#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void adagrad_update_1D_1D(float* x, float* d, float* m, float clip, float lr, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Using loop unrolling for performance improvement
    for (; tid < size; tid += stride * 4) {
        int index = tid;
        if (index < size) {
            float grad = d[index];
            grad = fminf(fmaxf(grad, -clip), clip); // Combine clipping in one line
            m[index] += grad * grad;
            x[index] -= lr * grad / sqrtf(m[index] + 1e-8f);
            d[index] = 0;
        }
        
        if (index + stride < size) {
            float grad = d[index + stride];
            grad = fminf(fmaxf(grad, -clip), clip);
            m[index + stride] += grad * grad;
            x[index + stride] -= lr * grad / sqrtf(m[index + stride] + 1e-8f);
            d[index + stride] = 0;
        }
        
        if (index + 2 * stride < size) {
            float grad = d[index + 2 * stride];
            grad = fminf(fmaxf(grad, -clip), clip);
            m[index + 2 * stride] += grad * grad;
            x[index + 2 * stride] -= lr * grad / sqrtf(m[index + 2 * stride] + 1e-8f);
            d[index + 2 * stride] = 0;
        }
        
        if (index + 3 * stride < size) {
            float grad = d[index + 3 * stride];
            grad = fminf(fmaxf(grad, -clip), clip);
            m[index + 3 * stride] += grad * grad;
            x[index + 3 * stride] -= lr * grad / sqrtf(m[index + 3 * stride] + 1e-8f);
            d[index + 3 * stride] = 0;
        }
    }
}