#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_sqrt(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll loop for better performance
    for (; tid < size; tid += stride * 4) {
        if (tid < size) out[tid] = sqrt(in[tid]);
        if (tid + stride < size) out[tid + stride] = sqrt(in[tid + stride]);
        if (tid + 2 * stride < size) out[tid + 2 * stride] = sqrt(in[tid + 2 * stride]);
        if (tid + 3 * stride < size) out[tid + 3 * stride] = sqrt(in[tid + 3 * stride]);
    }
}