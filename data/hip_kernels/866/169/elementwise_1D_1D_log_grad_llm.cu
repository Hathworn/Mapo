#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_log_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Unroll the loop for improved performance
    for (; tid + 3 < size; tid += 4 * stride) {
        in_d[tid] += out_d[tid] / in_x[tid];
        in_d[tid + stride] += out_d[tid + stride] / in_x[tid + stride];
        in_d[tid + 2 * stride] += out_d[tid + 2 * stride] / in_x[tid + 2 * stride];
        in_d[tid + 3 * stride] += out_d[tid + 3 * stride] / in_x[tid + 3 * stride];
    }
    for (; tid < size; tid += stride) { // Handle remaining elements
        in_d[tid] += out_d[tid] / in_x[tid];
    }
}