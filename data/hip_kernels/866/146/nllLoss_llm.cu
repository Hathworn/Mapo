#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nllLoss(float *x, int x_stride, float *y, int* target) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    // Check if tid is within bounds to prevent out-of-bounds access
    if (tid < gridDim.x * blockDim.x) {
        int offset = tid * x_stride + target[tid];
        y[tid] = -1.0f * x[offset];
    }
}