#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_add(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Simplified loop with unrolled if statement
    for (int i = tid; i < size; i += stride) {
        out[i] = in1[i] + in2[i];
    }
}