#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_mul_mutate(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Optimize loop to avoid unnecessary boundary check
    if (tid < size) {
        for (; tid < size; tid += stride) {
            out[tid] += in1[tid] * in2[tid];
        }
    }
}