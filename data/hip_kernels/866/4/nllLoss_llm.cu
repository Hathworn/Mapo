#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void nllLoss(float *x, int x_stride, float *y, int* target) {
    // Calculate a unique thread index for current thread
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Avoid accessing out of bounds
    if (tid < gridDim.x * blockDim.x) {
        // Calculate memory offset for the current thread
        int offset = __mul24(tid, x_stride) + target[tid];
        
        // Assign loss value to output array
        y[tid] = -x[offset];
    }
}