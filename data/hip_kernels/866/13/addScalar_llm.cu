#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void addScalar(float* in, float* out, float add, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop unrolling for performance improvement
    for (; tid + 3 < size; tid += stride * 4) {
        out[tid] = in[tid] + add;
        out[tid + 1] = in[tid + 1] + add;
        out[tid + 2] = in[tid + 2] + add;
        out[tid + 3] = in[tid + 3] + add;
    }

    // Handle remaining elements
    for (; tid < size; tid += stride) {
        out[tid] = in[tid] + add;
    }
}