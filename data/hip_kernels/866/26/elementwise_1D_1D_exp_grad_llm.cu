#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_exp_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use blockDim.x for better access pattern and performance
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        in_d[i] += out_d[i] * out_x[i];
    }
}