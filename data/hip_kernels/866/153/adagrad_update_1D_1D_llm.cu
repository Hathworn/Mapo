#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adagrad_update_1D_1D(float* x, float* d, float* m, float clip, float lr, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Use shared memory for faster access
    extern __shared__ float shared_d[];
    if (tid < size) shared_d[threadIdx.x] = d[tid];
    __syncthreads();

    for (; tid < size; tid += stride) {
        // Load data into registers for fast access
        float grad = shared_d[threadIdx.x];
        if (grad > clip) grad = clip;
        if (grad < -clip) grad = -clip;
        
        // Reduce global memory access by using local variable
        float momentum = m[tid] + grad * grad;
        m[tid] = momentum;

        float update = lr * grad / sqrtf(momentum + 1e-8f);
        x[tid] -= update;
        shared_d[threadIdx.x] = 0;  // Reset in shared memory
    }
}