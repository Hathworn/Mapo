#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void clipAt(float* in, float bound, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (; tid < size; tid += stride) {
        // Optimize boundary condition checks by removing redundant if statement
        float val = in[tid];
        if (val > bound) {
            in[tid] = bound;
        } else if (val < -bound) {
            in[tid] = -bound;
        }
    }
}