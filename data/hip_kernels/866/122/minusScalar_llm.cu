#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void minusScalar(float* in, float* out, float minus, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use a more efficient loop structure without if-check inside
    while (tid < size) {
        out[tid] = in[tid] - minus;
        tid += stride;
    }
}