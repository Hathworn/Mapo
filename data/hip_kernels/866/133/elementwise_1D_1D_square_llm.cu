#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwise_1D_1D_square(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a more compact loop by removing duplicate condition
    for (int index = tid; index < size; index += blockDim.x * gridDim.x) {
        out[index] = in[index] * in[index];
    }
}