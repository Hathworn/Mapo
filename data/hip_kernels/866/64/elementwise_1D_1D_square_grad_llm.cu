#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_square_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Optimize loop by removing the boundary check from the loop body
    for (; tid < size; tid += stride) {
        in_d[tid] += out_d[tid] * 2 * in_x[tid];
    }
}