#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_mul_mutate(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop unrolling to reduce iteration overhead
    for (; tid < size; tid += stride * 4) {
        if (tid < size) out[tid] += in1[tid] * in2[tid];
        if (tid + 1 < size) out[tid + 1] += in1[tid + 1] * in2[tid + 1];
        if (tid + 2 < size) out[tid + 2] += in1[tid + 2] * in2[tid + 2];
        if (tid + 3 < size) out[tid + 3] += in1[tid + 3] * in2[tid + 3];
    }
}