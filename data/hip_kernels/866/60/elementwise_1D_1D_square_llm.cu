#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_square(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop unrolling for performance optimization
    for (; tid < size; tid += stride * 4) {
        if (tid < size) out[tid] = in[tid] * in[tid];
        if (tid + stride < size) out[tid + stride] = in[tid + stride] * in[tid + stride];
        if (tid + 2 * stride < size) out[tid + 2 * stride] = in[tid + 2 * stride] * in[tid + 2 * stride];
        if (tid + 3 * stride < size) out[tid + 3 * stride] = in[tid + 3 * stride] * in[tid + 3 * stride];
    }
}