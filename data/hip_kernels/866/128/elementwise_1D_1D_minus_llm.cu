#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_minus(float* in1, float* in2, float* out, int size) {
    // Calculate global thread index and grid stride
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Optimize loop by removing redundant if condition
    for (; tid < size; tid += stride) {
        out[tid] = in1[tid] - in2[tid];
    }
}