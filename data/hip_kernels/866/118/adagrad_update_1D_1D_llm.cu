#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adagrad_update_1D_1D(float* x, float* d, float* m, float clip, float lr, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int index = tid; index < size; index += stride) {  // Use index variable to improve code clarity
        float grad = d[index];
        grad = fmaxf(-clip, fminf(clip, grad));  // Use fmaxf and fminf for clipping
        m[index] += grad * grad;
        x[index] -= lr * grad / sqrtf(m[index] + 1e-8f);  // Use sqrtf for single precision
        d[index] = 0.0f;  // Set to zero immediately after the update
    }
}