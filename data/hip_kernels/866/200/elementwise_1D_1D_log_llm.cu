#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_log(float* in, float* out, int size) {
    // Calculate thread ID using built-in variables
    int tid = blockIdx.x * blockDim.x + threadIdx.x; 
    int stride = gridDim.x * blockDim.x;

    // Loop over input elements with stride
    for (int i = tid; i < size; i += stride) {
        // Perform log calculation if index is within boundaries
        out[i] = logf(in[i]); // Use logf for better performance with float
    }
}