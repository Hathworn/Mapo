#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addScalarInArrayInPlace(float* in, float* add, float scale, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    float add_scaled = add[0] * scale; // Precompute scaled addition factor
    for (; tid < size; tid += stride) {
        // Remove redundant boundary check within loop
        in[tid] += add_scaled;
    }
}