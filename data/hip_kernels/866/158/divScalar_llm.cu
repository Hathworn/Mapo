#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void divScalar(float* in, float* out, float div, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Use tid for boundary check and computation
    if (tid < size) {
        out[tid] = in[tid] / div;
        tid += stride; // Increment tid for further loop iterations
    }
}