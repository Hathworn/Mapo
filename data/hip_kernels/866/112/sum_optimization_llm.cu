#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_optimization(float* in, int inStr0, int inStr1, int inStr2, int inStr3, float* out, int outStr0, int outStr1, int outStr2, int dim, int nElementOut, int dimSize) {
    // Compute thread id and stride
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Iterate over output elements with step size equal to stride
    for (int i = tid; i < nElementOut; i += stride) {
        // Calculate offsets for output indices
        int outOff0 = i / outStr0;
        int outOff1temp = i - outOff0 * outStr0;
        int outOff1 = outOff1temp / outStr1;
        int outOff2 = outOff1temp - outOff1 * outStr1;

        // Initialize sum accumulator
        float sum = 0.0f;

        // Loop over the dimension size and accumulate sum
        int baseOff = outOff0 * inStr0 + outOff1 * inStr1 + outOff2 * inStr2;
        for (int j = 0; j < dimSize; j++) {
            int inOff;
            if (dim == 0) inOff = j * inStr0 + outOff0 * inStr1 + outOff1 * inStr2 + outOff2 * inStr3;
            if (dim == 1) inOff = baseOff + j * inStr1;
            if (dim == 2) inOff = baseOff + j * inStr2;
            if (dim == 3) inOff = baseOff + j * inStr3;
            sum += in[inOff];
        }
        // Store accumulated sum into output
        out[i] = sum;
    }
}