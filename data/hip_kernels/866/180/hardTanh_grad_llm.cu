#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hardTanh_grad(float* in_x, float* in_d, float* out_d, float min_val, float max_val, int size, bool inplace) {
    // Calculate global index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure threads work on contiguous block of data
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        float x = in_x[i];
        if (inplace) {
            // Set derivative to zero for out-of-bound values
            in_d[i] = (x < min_val || x > max_val) ? 0 : in_d[i];
        } else {
            // Accumulate if value within bounds
            if (x >= min_val && x <= max_val) {
                in_d[i] += out_d[i];
            }
        }
    }
}