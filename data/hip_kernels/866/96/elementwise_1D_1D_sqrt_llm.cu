#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_sqrt(float* in, float* out, int size) {
    // Calculate thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Iterate over the elements using stride
    int stride = blockDim.x * gridDim.x;
    while (tid < size) {
        // Perform square root operation
        out[tid] = sqrtf(in[tid]);
        
        // Move to the next element in stride
        tid += stride;
    }
}