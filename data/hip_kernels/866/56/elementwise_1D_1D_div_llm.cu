#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_div(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    // Unrolling the loop to improve performance
    for (int i = tid; i < size; i += stride * 4) {
        if (i < size) out[i] = in1[i] / in2[i];
        if ((i + stride) < size) out[i + stride] = in1[i + stride] / in2[i + stride];
        if ((i + 2 * stride) < size) out[i + 2 * stride] = in1[i + 2 * stride] / in2[i + 2 * stride];
        if ((i + 3 * stride) < size) out[i + 3 * stride] = in1[i + 3 * stride] / in2[i + 3 * stride];
    }
}