#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_log(float* in, float* out, int size) {
    // Compute unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride loop to cover all elements
    int stride = blockDim.x * gridDim.x; 
    for (int i = tid; i < size; i += stride) {
        out[i] = log(in[i]); // Apply log operation
    }
}