#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addScalar(float* in, float* out, float add, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use modulo operation to ensure tid is within bounds
    if (tid < size) {
        out[tid] = in[tid] + add;
    }

    // Loop unrolling for better performance
    for (tid += stride; tid < size; tid += stride) {
        out[tid] = in[tid] + add;
    }
}