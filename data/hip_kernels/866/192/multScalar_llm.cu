#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multScalar(float* in, float* out, float mult, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; 
    int stride = gridDim.x * blockDim.x;
    // Optimize: Remove redundant check within the loop
    if (tid < size) {
        for (; tid < size; tid += stride) {
            out[tid] = in[tid] * mult;
        }
    }
}