#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_sqrt(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Optimize loop to remove condition inside and enhance warp execution
    while (tid < size) {
        out[tid] = sqrtf(in[tid]);  // Use single-precision sqrt function
        tid += stride;
    }
}