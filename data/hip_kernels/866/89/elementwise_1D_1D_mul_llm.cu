#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwise_1D_1D_mul(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Loop unrolling for better efficiency
    while (tid < size) {
        out[tid] = in1[tid] * in2[tid];
        tid += stride;
    }
}