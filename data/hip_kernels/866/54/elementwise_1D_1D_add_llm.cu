#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_add(float* in1, float* in2, float* out, int size) {
    // Calculate global index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Unrolling loop for better performance
    int stride = gridDim.x * blockDim.x * 4;
    for (; tid < size; tid += stride) {
        if (tid < size) out[tid] = in1[tid] + in2[tid];
        if (tid + blockDim.x < size) out[tid + blockDim.x] = in1[tid + blockDim.x] + in2[tid + blockDim.x];
        if (tid + 2 * blockDim.x < size) out[tid + 2 * blockDim.x] = in1[tid + 2 * blockDim.x] + in2[tid + 2 * blockDim.x];
        if (tid + 3 * blockDim.x < size) out[tid + 3 * blockDim.x] = in1[tid + 3 * blockDim.x] + in2[tid + 3 * blockDim.x];
    }
}