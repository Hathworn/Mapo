#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_exp_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
    // Use shared memory to reduce global memory accesses
    extern __shared__ float shared_out_x[];
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Load data into shared memory
    if (threadIdx.x < size) {
        shared_out_x[threadIdx.x] = out_x[threadIdx.x + blockIdx.x * blockDim.x];
    }
    
    __syncthreads(); // Synchronize to ensure shared memory is fully loaded

    for (; tid < size; tid += stride) {
        // Only proceed if within boundaries
        if (tid < size) {
            // Use shared memory for reading out_x
            in_d[tid] += out_d[tid] * shared_out_x[tid % blockDim.x];
        }
    }
}