#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_minus(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll the loop to reduce iterations and increase parallel execution
    for (int i = tid; i < size; i += stride * 4) {
        if (i < size) out[i] = in1[i] - in2[i];
        if (i + 1 < size) out[i + 1] = in1[i + 1] - in2[i + 1];
        if (i + 2 < size) out[i + 2] = in1[i + 2] - in2[i + 2];
        if (i + 3 < size) out[i + 3] = in1[i + 3] - in2[i + 3];
    }
}