#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask4D(float* in, int* mask, int xstrides0, int xstrides1, int xstrides2, int xstrides3, int scalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    while (tid < scalarCount) {
        int xindex0, xindex1, xindex2, xindex3;
        
        // Directly calculate indices from tid
        xindex0 = tid / xstrides0;
        tid = tid % xstrides0;
        xindex1 = tid / xstrides1;
        tid = tid % xstrides1;
        xindex2 = tid / xstrides2;
        xindex3 = tid % xstrides2;
        
        // Efficient mask check
        if (xindex3 >= mask[xindex0]) {
            in[tid] = 0;
        }

        // Update tid for the next iteration
        tid += stride;
    }
}