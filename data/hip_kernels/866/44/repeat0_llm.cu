#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void repeat0(float* __restrict__ in, float* __restrict__ out, int outStride0, int outStride1, int outScalarCount) {
    // Use shared memory to reduce global memory access latency
    extern __shared__ float sharedIn[];
    
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Each thread loads data into shared memory
    if (threadIdx.x < outScalarCount) {
        sharedIn[threadIdx.x] = in[threadIdx.x];
    }
    __syncthreads();

    for (; tid < outScalarCount; tid += stride) {
        int linearIndex = tid;
        int outIndex0 = linearIndex / outStride0;
        linearIndex = linearIndex - outIndex0 * outStride0;
        int outIndex1 = linearIndex / outStride1;
        int outIndex2 = linearIndex - outIndex1 * outStride1;
        
        // Use shared memory in place of global memory access
        int inIndex = outIndex2 + (outIndex0 + outIndex1) * outStride1;
        out[tid] = sharedIn[inIndex];
    }
}