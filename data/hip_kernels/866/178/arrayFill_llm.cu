#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void arrayFill(float* data, float value, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll the loop for better performance
    for (int i = tid; i < size; i += stride * 4) {
        data[i] = value;
        if (i + stride < size) data[i + stride] = value;
        if (i + stride * 2 < size) data[i + stride * 2] = value;
        if (i + stride * 3 < size) data[i + stride * 3] = value;
    }
}