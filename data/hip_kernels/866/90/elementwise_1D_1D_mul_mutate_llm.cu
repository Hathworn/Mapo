#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_mul_mutate(float* in1, float* in2, float* out, int size) {
    // Calculate thread id once
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Adjust loop to avoid boundary checks in each iteration
    for (; tid < size; tid += stride) {
        out[tid] += in1[tid] * in2[tid];
    }
}