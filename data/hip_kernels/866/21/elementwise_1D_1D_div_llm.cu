#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_div(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use while loop for clarity and ensure stride coverage
    while (tid < size) {
        out[tid] = in1[tid] / in2[tid];  // Perform element-wise division
        tid += stride;
    }
}