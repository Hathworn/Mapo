#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addScalarInArrayInPlace(float* in, float* add, float scale, int size) {
    // Calculate unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride to ensure all blocks/threads process the entire array
    int stride = blockDim.x * gridDim.x;

    // Loop through array with strides, no need to check 'tid < size' inside loop
    while (tid < size) {
        in[tid] += add[0] * scale;  // Direct scaling
        tid += stride;
    }
}