#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void divScalar(float* in, float* out, float div, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Reduce branch divergence
    if (tid < size) {
        for (; tid < size; tid += stride) {
            out[tid] = in[tid] / div;
        }
    }
}