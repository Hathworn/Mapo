#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addScalar(float* in, float* out, float add, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Loop unrolling for improved performance
    for (; tid < size; tid += stride) {
        if (tid < size) out[tid] = in[tid] + add;
        if (tid + stride < size) out[tid + stride] = in[tid + stride] + add;
    }
}