#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_square(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use a while loop for cleaner logic
    while (tid < size) {
        out[tid] = in[tid] * in[tid];
        tid += stride;
    }
}