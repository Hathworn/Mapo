#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_square(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Use unrolled loop to optimize memory access
    for (; tid + 3 < size; tid += stride * 4) {
        out[tid] = in[tid] * in[tid];
        out[tid + 1] = in[tid + 1] * in[tid + 1];
        out[tid + 2] = in[tid + 2] * in[tid + 2];
        out[tid + 3] = in[tid + 3] * in[tid + 3];
    }
    // Handle remaining elements
    for (; tid < size; tid += stride) {
        out[tid] = in[tid] * in[tid];
    }
}