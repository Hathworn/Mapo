#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void adagrad_update_1D_1D(float* x, float* d, float* m, float clip, float lr, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Optimize: Use shared memory for accessing "clip" and "lr"
    __shared__ float sharedClip, sharedLr;
    if (threadIdx.x == 0) {
        sharedClip = clip;
        sharedLr = lr;
    }
    __syncthreads();

    for (; tid < size; tid += stride) {
        // Use sharedClip and sharedLr instead of global memory access
        if (d[tid] > sharedClip) d[tid] = sharedClip;
        if (d[tid] < -sharedClip) d[tid] = -sharedClip;
        m[tid] += d[tid] * d[tid];
        x[tid] -= sharedLr * d[tid] / sqrt(m[tid] + 0.00000001);
        d[tid] = 0;
    }
}