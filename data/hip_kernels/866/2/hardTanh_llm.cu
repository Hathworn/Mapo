#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void hardTanh(float* in, float* out, float min_val, float max_val, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = tid; i < size; i += stride) {
        // Use clamp function for better readability and possible performance improvement
        out[i] = fmaxf(min_val, fminf(max_val, in[i]));
    }
}