#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_sqrt_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Improved the loop condition to remove unnecessary check
    int stride = gridDim.x * blockDim.x;

    while (tid < size) {
        in_d[tid] += out_d[tid] / out_x[tid] / 2;
        tid += stride; // Increment tid with stride inside the loop to prevent multiple calculations
    }
}