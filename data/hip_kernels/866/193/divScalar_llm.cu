#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void divScalar(float* in, float* out, float div, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Loop unrolling for performance improvement
    for (; tid + 3 < size; tid += stride * 4) {
        out[tid] = in[tid] / div;
        out[tid + 1] = in[tid + 1] / div;
        out[tid + 2] = in[tid + 2] / div;
        out[tid + 3] = in[tid + 3] / div;
    }
    // Handle remaining elements
    for (; tid < size; tid += stride) {
        out[tid] = in[tid] / div;
    }
}