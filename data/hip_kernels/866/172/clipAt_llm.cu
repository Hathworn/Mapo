#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void clipAt(float* in, float bound, int size) {
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure each thread processes elements spaced by total number of threads
    for (; tid < size; tid += blockDim.x * gridDim.x) {
        // Clip values at bounds
        if (in[tid] > bound) in[tid] = bound;
        else if (in[tid] < -bound) in[tid] = -bound;
    }
}