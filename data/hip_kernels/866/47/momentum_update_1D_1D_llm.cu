#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void momentum_update_1D_1D(float* x, float* d, float* m, float learning_rate, float momentum, float gradClip, bool nesterov, int size) {
    // Efficient tid calculation to minimize warp divergence
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Using shared memory to reduce global memory access; not applicable directly here, maintain simple optimization
    for (; tid < size; tid += stride) {
        float temp = d[tid];

        // Use ternary operator for compact and potentially faster comparison and assignment
        temp = temp > gradClip ? gradClip : (temp < -gradClip ? -gradClip : temp);

        // Minimize operations by reducing multiple accesses
        float m_tid = m[tid] * momentum + temp;
        m[tid] = m_tid;

        // Conditional logic optimized to minimize branch divergence
        temp = nesterov ? temp + momentum * m_tid : m_tid;

        // Update x array with efficient access
        x[tid] -= learning_rate * temp;

        // Reset d array to zero; maintain efficient writing
        d[tid] = 0;
    }
}