#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_square(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop unrolling for efficiency
    for (; tid + 3 < size; tid += 4 * stride) {
        out[tid] = in[tid] * in[tid];
        out[tid + stride] = in[tid + stride] * in[tid + stride];
        out[tid + 2 * stride] = in[tid + 2 * stride] * in[tid + 2 * stride];
        out[tid + 3 * stride] = in[tid + 3 * stride] * in[tid + 3 * stride];
    }

    // Process any remaining elements
    while (tid < size) {
        out[tid] = in[tid] * in[tid];
        tid += stride;
    }
}