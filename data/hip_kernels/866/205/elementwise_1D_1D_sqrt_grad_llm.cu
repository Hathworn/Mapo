#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_sqrt_grad(float* in_x, float* in_d, float* out_x, float* out_d, int size) {
    // Calculate unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride for efficient memory access pattern
    int stride = blockDim.x * gridDim.x;

    // Process elements in stride increments
    while (tid < size) {
        in_d[tid] += out_d[tid] / (2 * out_x[tid]); // Optimize division and operation order
        tid += stride;
    }
}