#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_exp(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = gridDim.x * blockDim.x; // Precompute the total number of threads

    for (; tid < size; tid += gridSize) {
        // Perform the operation only when within bounds
        out[tid] = exp(in[tid]);
    }
}