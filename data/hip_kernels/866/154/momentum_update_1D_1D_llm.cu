#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void momentum_update_1D_1D(float* x, float* d, float* m, float learning_rate, float momentum, float gradClip, bool nesterov, int size) {
    // Calculate thread id and stride
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (; tid < size; tid += stride) {
        float temp = d[tid];
        
        // Apply gradient clipping
        temp = max(min(temp, gradClip), -gradClip);

        // Update momentum
        float m_val = m[tid] * momentum + temp;

        // Update parameter using Nesterov or standard method
        x[tid] -= learning_rate * (nesterov ? temp + momentum * m_val : m_val);

        // Store updated momentum and reset gradient
        m[tid] = m_val;
        d[tid] = 0;
    }
}