#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_exp_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a grid-stride loop for better resource utilization
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        in_d[i] += out_d[i] * out_x[i];
    }
}