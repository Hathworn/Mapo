#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void repeat0(float* in, float* out, int outStride0, int outStride1, int outScalarCount) {
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    while (tid < outScalarCount) {
        // Simplify linear index calculation using div and mod operations for better readability and performance
        int outIndex0 = tid / outStride0;
        tid = tid % outStride0;
        int outIndex1 = tid / outStride1;
        int outIndex2 = tid % outStride1;

        // Calculate input index with reduced arithmetic operations
        int inIndex = outIndex2 + outIndex1 * outStride1 + outIndex0 * outStride1;

        // Write output
        out[tid] = in[inIndex];

        // Advance to the next index
        tid += stride;
    }
}