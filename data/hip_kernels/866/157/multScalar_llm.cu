#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multScalar(float* in, float* out, float mult, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Loop unrolling for better performance
    for (int i = tid; i < size; i += stride * 4) {
        if (i < size) out[i] = in[i] * mult;
        if (i + stride < size) out[i + stride] = in[i + stride] * mult;
        if (i + 2 * stride < size) out[i + 2 * stride] = in[i + 2 * stride] * mult;
        if (i + 3 * stride < size) out[i + 3 * stride] = in[i + 3 * stride] * mult;
    }
}