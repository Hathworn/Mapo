#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addScalar(float* in, float* out, float add, int size) {
    // Calculate the thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Improved loop with tid check within loop condition
    for (; tid < size; tid += stride) {
        out[tid] = in[tid] + add;
    }
}