#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void momentum_update_1D_1D(float* x, float* d, float* m, float learning_rate, float momentum, float gradClip, bool nesterov, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use __restrict__ to indicate no aliasing
    float* __restrict__ d_restrict = d;
    float* __restrict__ m_restrict = m;
    
    for (; tid < size; tid += stride) {
        // Efficient memory access
        float temp = __ldg(&d_restrict[tid]);
        
        // Apply gradient clipping
        if (temp > gradClip) temp = gradClip;
        if (temp < -gradClip) temp = -gradClip;
        
        // Update momentum
        float* m_ptr = &m_restrict[tid];
        *m_ptr = *m_ptr * momentum + temp;
        
        // Apply Nesterov acceleration if enabled
        temp = nesterov ? (temp + momentum * *m_ptr) : *m_ptr;
        
        // Update position vector
        x[tid] -= learning_rate * temp;
        
        // Reset gradient
        d_restrict[tid] = 0.0f;
    }
}