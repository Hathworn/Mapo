#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_square_grad(float* in_x, float* in_d, float* out_x, float* out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (; tid < size; tid += stride) {
        // Optimize: Use register to store repeated calculation
        float temp = out_d[tid] * 2 * in_x[tid];
        in_d[tid] += temp;
    }
}