#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void mul_sub_grad(float* in1_x, float* in1_d, float* in2_x, float* in2_d, float* out, int in1ScalarCount, int in2ScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Pre-calculate commonly used values
    while (tid < in1ScalarCount) {
        int index = tid % in2ScalarCount;
        
        // Optimize memory access patterns
        float out_val = out[tid];
        float in2_x_val = in2_x[index];
        float in1_x_val = in1_x[tid];
        
        // Shared memory can be used if further optimization is needed depending on in2ScalarCount size
        in1_d[tid] += out_val * in2_x_val;
        in2_d[tid] = in1_x_val * out_val;  // requires reduction

        tid += stride;
    }
}