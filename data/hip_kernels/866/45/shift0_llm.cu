#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shift0(float* in, float* out, int inDim0, int inStride0, int inStride1, int inScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use shared memory for better memory access pattern if data reuse is possible (not utilized here but an idea to consider)
    // __shared__ float sharedMem[BLOCK_SIZE];

    while (tid < inScalarCount) { // Change for loop to while loop for potential readability and slight overhead reduction
        int linearIndex = tid;
        int inIndex0 = linearIndex / inStride0;
        linearIndex -= inIndex0 * inStride0; // Use compound assignment for clarity
        int inIndex1 = linearIndex / inStride1;

        // Remove if condition and use mask to ensure 'out' assignment only when the condition is met
        // This aids warp efficiency by avoiding divergent branches
        bool valid = (inIndex0 + inIndex1 < inDim0);
        out[tid + inIndex1 * inStride0] = valid ? in[tid] : 0.0f;

        tid += stride;
    }
}