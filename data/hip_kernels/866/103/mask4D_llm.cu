#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mask4D(float* in, int* mask, int xstrides0, int xstrides1, int xstrides2, int xstrides3, int scalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll loop for better performance
    for (; tid < scalarCount; tid += stride) {
        // Inline calculations using modulo to avoid repeated division
        int xindex0 = tid / xstrides0;
        int remainder0 = tid % xstrides0;
        int xindex1 = remainder0 / xstrides1;
        int remainder1 = remainder0 % xstrides1;
        int xindex2 = remainder1 / xstrides2;
        int xindex3 = remainder1 % xstrides2;

        // Simplified conditional checking
        if (xindex3 >= mask[xindex0]) 
            in[tid] = 0;
    }
}