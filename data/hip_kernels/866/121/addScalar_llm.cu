#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addScalar(float* in, float* out, float add, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (int i = tid; i < size; i += stride) { // Use local variable for loop control
        out[i] = in[i] + add; // Calculate and store result directly
    }
}