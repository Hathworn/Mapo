#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void arrayFill(float* data, float value, int size) {
    // Cache grid and block calculations in registers
    int stride = gridDim.x * blockDim.x;
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Use __restrict__ keyword for data pointer to enhance compiler optimization
    float* __restrict__ data_ptr = data;

    for (int i = tid; i < size; i += stride) {
        // Ensure coalesced memory access pattern if possible
        data_ptr[i] = value;
    }
}