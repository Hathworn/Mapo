#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_div(float* in1, float* in2, float* out, int size) {
    // Improve memory access pattern using shared memory.
    extern __shared__ float shared_data[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < size) {
        // Load data into shared memory from global memory.
        shared_data[threadIdx.x] = in2[tid];
        __syncthreads();

        // Perform division if denominator is non-zero to avoid undefined behavior.
        float denominator = shared_data[threadIdx.x];
        out[tid] = denominator != 0.0f ? in1[tid] / denominator : 0.0f; 
    }
}