#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nllLoss(float *x, int x_stride, float *y, int* target) {
    // Use a shared memory to reduce redundant calculations
    extern __shared__ float shared_x[];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int lane = threadIdx.x;  // local thread index in the block

    // Load relevant data into shared memory
    shared_x[lane] = x[tid * x_stride + target[tid]];
    __syncthreads();

    // Perform the loss calculation
    y[tid] = -1.0f * shared_x[lane];
}