#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mul_sub(float* in1, float* in2, float* out, int in1ScalarCount, int in2ScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Optimize by unrolling the loop for improved performance
    int tidMod = tid % in2ScalarCount;
    for (; tid < in1ScalarCount; tid += stride) {
        out[tid] = in1[tid] * in2[tidMod];
        tidMod = (tidMod + 1) % in2ScalarCount; // Calculate mod outside the loop
    }
}