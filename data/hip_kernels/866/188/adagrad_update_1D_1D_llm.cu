#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adagrad_update_1D_1D(float* x, float* d, float* m, float clip, float lr, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Use a while loop for better readability and structure
    while (tid < size) {
        // Clip gradients more efficiently using fminf/fmaxf
        d[tid] = fmaxf(fminf(d[tid], clip), -clip);

        // Update accumulator and parameters
        m[tid] += d[tid] * d[tid];
        x[tid] -= lr * d[tid] / sqrtf(m[tid] + 1e-8f);

        // Reset gradient
        d[tid] = 0;

        tid += stride;
    }
}