#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shift0(float* in, float* out, int inDim0, int inStride0, int inStride1, int inScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use loop unrolling for better performance
    for (; tid < inScalarCount; tid += stride) {
        int linearIndex = tid;
        int inIndex0 = linearIndex / inStride0;
        linearIndex -= inIndex0 * inStride0;
        int inIndex1 = linearIndex / inStride1;

        // Improve branch prediction by rearranging condition logic
        if (inIndex0 + inIndex1 < inDim0) {
            out[tid + inIndex1 * inStride0] = in[tid];
        }
    }
}