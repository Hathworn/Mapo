#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void clipAt(float* in, float bound, int size) {
    // Calculate unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if index is within bounds and apply bounds clipping
    if (tid < size) {
        float value = in[tid];
        in[tid] = value > bound ? bound : (value < -bound ? -bound : value);
    }
}