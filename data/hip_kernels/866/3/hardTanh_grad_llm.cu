#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void hardTanh_grad(float* in_x, float* in_d, float* out_d, float min_val, float max_val, int size, bool inplace) {
    // Calculate global thread ID for better memory access pattern
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Unroll loop for better performance
    for (int i = tid; i < size; i += stride) {
        float x = in_x[i]; // Avoid repeated global memory access
        if (inplace) {
            if (x < min_val || x > max_val) {
                in_d[i] = 0; // Prevent redundant operations
            }
        } else {
            if (x >= min_val && x <= max_val) {
                in_d[i] += out_d[i]; // Avoid redundant operations
            }
        }
    }
}