#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_sub(float* in1, float* in2, float* out, int in1ScalarCount, int in2ScalarCount) {
    // Using blockIdx.x, blockDim.x, and threadIdx.x to calculate unique thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Precompute grid stride
    int stride = blockDim.x * gridDim.x;
    
    // Use a grid-stride loop for better occupancy and workload distribution
    while (tid < in1ScalarCount) {
        out[tid] = in1[tid] * in2[tid % in2ScalarCount];
        tid += stride;
    }
}