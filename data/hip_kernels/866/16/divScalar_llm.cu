#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void divScalar(float* __restrict__ in, float* __restrict__ out, float div, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Use loop unrolling for performance improvement
    for (; tid < size; tid += stride * 4) {
        if (tid < size) out[tid] = in[tid] / div;
        if (tid + stride < size) out[tid + stride] = in[tid + stride] / div;
        if (tid + 2 * stride < size) out[tid + 2 * stride] = in[tid + 2 * stride] / div;
        if (tid + 3 * stride < size) out[tid + 3 * stride] = in[tid + 3 * stride] / div;
    }
}