#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void minusScalar(float* in, float* out, float minus, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop with consistent boundary check
    while (tid < size) {
        out[tid] = in[tid] - minus; // Perform operation
        tid += stride;
    }
}