#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_mul_mutate(float* in1, float* in2, float* out, int size) {
    // Simplify thread index calculation using a single variable
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a more efficient loop condition
    if (tid < size) {
        out[tid] += in1[tid] * in2[tid];
    }
}