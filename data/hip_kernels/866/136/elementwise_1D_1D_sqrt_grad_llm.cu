#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_sqrt_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Optimize by removing branch condition from the loop
    if (tid < size) {
        int stride = gridDim.x * blockDim.x;
        // Unroll the loop to process multiple elements per thread
        for (; tid < size; tid += stride) {
            in_d[tid] += out_d[tid] / out_x[tid] / 2;
        }
    }
}