#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void shift0(float* in, float* out, int inDim0, int inStride0, int inStride1, int inScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use a single loop and eliminate redundant calculations
    while (tid < inScalarCount) {
        int inIndex0 = tid / inStride0;
        int inIndex1 = (tid % inStride0) / inStride1;
        
        if (inIndex0 + inIndex1 < inDim0) {
            // Ensure only valid indices are processed
            out[tid + inIndex1 * inStride0] = in[tid];
        }
        
        tid += stride; // Increment tid by stride for next iteration
    }
}