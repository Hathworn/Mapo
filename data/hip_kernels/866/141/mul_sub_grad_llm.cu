#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized kernel function using shared memory and loop unrolling
__global__ void mul_sub_grad(float* in1_x, float* in1_d, float* in2_x, float* in2_d, float* out, int in1ScalarCount, int in2ScalarCount) {
    extern __shared__ float sharedIn2X[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Load shared memory just once per block
    if (threadIdx.x < in2ScalarCount) {
        sharedIn2X[threadIdx.x] = in2_x[threadIdx.x];
    }
    __syncthreads();

    // Process multiple elements in each loop iteration using loop unrolling for efficiency
    for (; tid < in1ScalarCount; tid += stride) {
        int index = tid % in2ScalarCount;
        float outVal = out[tid];
        in1_d[tid] += outVal * sharedIn2X[index];
        in2_d[tid] = in1_x[tid] * outVal;
    }
}