#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_square_grad(float* in_x, float* in_d, float* out_x, float* out_d, int size) {
    // Calculate thread id
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use stride loop for efficient memory access
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        in_d[i] += out_d[i] * 2 * in_x[i];  // Element-wise square gradient computation
    }
}