#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask4D(float* in, int* mask, int xstrides0, int xstrides1, int xstrides2, int xstrides3, int scalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    while (tid < scalarCount) {
        // Calculate indices using fast integer division and modulus
        int xindex0 = tid / xstrides0;
        int rem0 = tid % xstrides0;
        int xindex1 = rem0 / xstrides1;
        int rem1 = rem0 % xstrides1;
        int xindex2 = rem1 / xstrides2;
        int xindex3 = rem1 % xstrides2;

        // Avoid repeated access to mask for optimization
        int threshold = mask[xindex0];
        if (xindex3 >= threshold) {
            in[tid] = 0;
        }

        tid += stride;
    }
}