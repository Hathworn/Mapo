#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_exp_grad(float* in_x, float* in_d, float* out_x, float* out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Unroll loop to increase memory access throughput
    for (int i = tid; i < size; i += stride * 4) {
        if (i < size) in_d[i] += out_d[i] * out_x[i];
        if (i + stride < size) in_d[i + stride] += out_d[i + stride] * out_x[i + stride];
        if (i + 2 * stride < size) in_d[i + 2 * stride] += out_d[i + 2 * stride] * out_x[i + 2 * stride];
        if (i + 3 * stride < size) in_d[i + 3 * stride] += out_d[i + 3 * stride] * out_x[i + 3 * stride];
    }
}