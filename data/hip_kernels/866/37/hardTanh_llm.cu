#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hardTanh(float* in, float* out, float min_val, float max_val, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    // Optimize by unrolling the loop for better performance
    #pragma unroll
    for (int i = tid; i < size; i += stride) {
        float val = in[i];
        val = val < min_val ? min_val : val;
        val = val > max_val ? max_val : val;
        out[i] = val;
    }
}