#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwise_1D_1D_exp(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Ensure tid starts within bounds and use stride efficiently
    while (tid < size) {
        out[tid] = exp(in[tid]);
        tid += stride;
    }
}