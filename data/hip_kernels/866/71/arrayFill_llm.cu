#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void arrayFill(float* data, float value, int size) {
    // Calculate thread ID and stride
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    
    // Loop iterates over elements
    for (int i = tid; i < size; i += stride) {
        data[i] = value; // Assign value to the current element
    }
}