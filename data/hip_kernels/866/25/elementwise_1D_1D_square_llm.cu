#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_square(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop and process elements with stride, no need for boundary check inside loop
    while (tid < size) {
        out[tid] = in[tid] * in[tid];
        tid += stride;
    }
}