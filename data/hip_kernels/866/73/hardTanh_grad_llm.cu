#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hardTanh_grad(float* in_x, float* in_d, float* out_d, float min_val, float max_val, int size, bool inplace) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    // Use shared memory to optimize memory access pattern
    __shared__ float shared_in[1024];
    __shared__ float shared_out[1024];
    
    for (int i = tid; i < size; i += stride) {
        // Load data into shared memory
        shared_in[threadIdx.x] = in_x[i];
        shared_out[threadIdx.x] = out_d[i];
        __syncthreads();

        if (inplace) {
            // Direct update for in-place
            if (shared_in[threadIdx.x] < min_val || shared_in[threadIdx.x] > max_val)
                in_d[i] = 0;
        } else {
            // Accumulate only within the specified range
            if (shared_in[threadIdx.x] >= min_val && shared_in[threadIdx.x] <= max_val)
                in_d[i] += shared_out[threadIdx.x];
        }
        __syncthreads();
    }
}