#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_sub(float* in1, float* in2, float* out, int in1ScalarCount, int in2ScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    int in2Idx;  // Declare variable outside loop for reused index calculation

    // Use loop unrolling for improved performance
    for (; tid < in1ScalarCount; tid += stride * 4) {
        in2Idx = tid % in2ScalarCount;
        if (tid < in1ScalarCount) 
            out[tid] = in1[tid] * in2[in2Idx];
        if (tid + stride < in1ScalarCount) 
            out[tid + stride] = in1[tid + stride] * in2[(tid + stride) % in2ScalarCount];
        if (tid + 2 * stride < in1ScalarCount) 
            out[tid + 2 * stride] = in1[tid + 2 * stride] * in2[(tid + 2 * stride) % in2ScalarCount];
        if (tid + 3 * stride < in1ScalarCount) 
            out[tid + 3 * stride] = in1[tid + 3 * stride] * in2[(tid + 3 * stride) % in2ScalarCount];
    }
}