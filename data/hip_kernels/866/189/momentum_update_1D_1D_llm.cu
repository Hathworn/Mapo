#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void momentum_update_1D_1D(float* x, float* d, float* m, float learning_rate, float momentum, float gradClip, bool nesterov, int size) {
    // Cache stride calculation
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Loop unrolling for better performance
    for (int i = tid; i < size; i += stride) {
        // Efficient clamping using min/max functions
        float temp = fmaxf(fminf(d[i], gradClip), -gradClip);

        // Momentum calculation
        float m_update = momentum * m[i] + temp;
        if (nesterov) {
            temp += momentum * m_update;
        } else {
            temp = m_update;
        }

        // Efficiently update values
        m[i] = m_update;
        x[i] -= learning_rate * temp;
        d[i] = 0; // Reset gradient
    }
}