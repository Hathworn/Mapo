#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void mask4D(float* in, int* mask, int xstrides0, int xstrides1, int xstrides2, int xstrides3, int scalarCount) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use shared memory to load mask values to reduce global memory accesses
    __shared__ int shared_mask[32];  // Assuming xstrides0 <= 32, adjust if necessary
    if(threadIdx.x < xstrides0) {
        shared_mask[threadIdx.x] = mask[threadIdx.x];
    }
    __syncthreads();

    for (; tid < scalarCount; tid += stride) {
        int linearIndex = tid;
        int xindex0 = linearIndex / xstrides0;
        linearIndex -= xstrides0 * xindex0;
        int xindex1 = linearIndex / xstrides1;
        linearIndex -= xstrides1 * xindex1;
        int xindex2 = linearIndex / xstrides2;
        int xindex3 = linearIndex - xstrides2 * xindex2;
        
        // Use shared memory reference for mask
        if (xindex3 >= shared_mask[xindex0]) in[tid] = 0;
    }
}