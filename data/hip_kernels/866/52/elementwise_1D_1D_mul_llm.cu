#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_mul(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop unrolling for better performance
    for (int i = tid; i < size; i += stride * 4) {
        if (i < size) out[i] = in1[i] * in2[i];
        if (i + stride < size) out[i + stride] = in1[i + stride] * in2[i + stride];
        if (i + stride * 2 < size) out[i + stride * 2] = in1[i + stride * 2] * in2[i + stride * 2];
        if (i + stride * 3 < size) out[i + stride * 3] = in1[i + stride * 3] * in2[i + stride * 3];
    }
}