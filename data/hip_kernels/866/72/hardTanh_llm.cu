#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hardTanh(float* in, float* out, float min_val, float max_val, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    // Use shared memory for faster access
    extern __shared__ float shared_in[];
    shared_in[threadIdx.x] = in[tid];
    __syncthreads();

    for (int i = tid; i < size; i += stride) {
        float val = shared_in[threadIdx.x];
        out[i] = val < min_val ? min_val : (val > max_val ? max_val : val);
    }
}