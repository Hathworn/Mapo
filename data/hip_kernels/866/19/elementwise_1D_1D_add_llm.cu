#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_add(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique thread index
    int stride = blockDim.x * gridDim.x; // Total threads in the grid
    while (tid < size) { // Efficient loop for processing data
        out[tid] = in1[tid] + in2[tid]; // Perform element-wise addition
        tid += stride; // Move to the next element handled by the thread
    }
}