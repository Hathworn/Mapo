#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void nllLoss_grad(int x_stride, float *yGrad, int* target, float* xGrad) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Calculate memory offset and update gradient if within range
    if (tid < gridDim.x * blockDim.x) {
        int offset = tid * x_stride + target[tid];
        atomicAdd(&xGrad[offset], -yGrad[tid]);  // Use atomic operation for safe updates
    }
}