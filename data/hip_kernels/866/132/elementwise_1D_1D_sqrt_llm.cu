#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwise_1D_1D_sqrt(float* in, float* out, int size) {
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a while loop for stride access which is more efficient
    while (tid < size) {
        out[tid] = sqrtf(in[tid]); // Use sqrtf for float inputs
        tid += gridDim.x * blockDim.x;
    }
}