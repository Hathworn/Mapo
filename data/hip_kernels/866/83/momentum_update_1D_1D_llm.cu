#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void momentum_update_1D_1D(float* x, float* d, float* m, float learning_rate, float momentum, float gradClip, bool nesterov, int size) {
    // Optimize kernel with loop unrolling for better performance
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    #pragma unroll 4
    for (; tid < size; tid += stride) {
        float temp = d[tid];
        
        // Apply gradient clipping
        temp = fminf(fmaxf(temp, -gradClip), gradClip);
        
        // Update momentum
        m[tid] = momentum * m[tid] + temp;
        
        // Apply Nesterov momentum if needed
        if (nesterov) { 
            temp += momentum * m[tid]; 
        } else { 
            temp = m[tid]; 
        }
        
        // Update parameters
        x[tid] -= learning_rate * temp;
        d[tid] = 0;
    }
}