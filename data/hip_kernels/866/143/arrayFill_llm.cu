#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void arrayFill(float* data, float value, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    
    // Loop unrolling for increased performance
    for (int i = tid; i < size; i += 4 * stride) {
        if (i < size) data[i] = value;
        if (i + stride < size) data[i + stride] = value;
        if (i + 2 * stride < size) data[i + 2 * stride] = value;
        if (i + 3 * stride < size) data[i + 3 * stride] = value;
    }
}