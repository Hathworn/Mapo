#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void clipAt(float* in, float bound, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll loop to reduce iterations and improve performance
    for (int i = tid; i < size; i += stride * 4) {
        if (i < size) {
            if (in[i] > bound) in[i] = bound;
            if (in[i] < -bound) in[i] = -bound;
        }
        if (i + stride < size) {
            if (in[i + stride] > bound) in[i + stride] = bound;
            if (in[i + stride] < -bound) in[i + stride] = -bound;
        }
        if (i + stride * 2 < size) {
            if (in[i + stride * 2] > bound) in[i + stride * 2] = bound;
            if (in[i + stride * 2] < -bound) in[i + stride * 2] = -bound;
        }
        if (i + stride * 3 < size) {
            if (in[i + stride * 3] > bound) in[i + stride * 3] = bound;
            if (in[i + stride * 3] < -bound) in[i + stride * 3] = -bound;
        }
    }
}