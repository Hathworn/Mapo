#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_sqrt(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure threads only operate within valid size range
    if (tid < size) {
        out[tid] = sqrt(in[tid]);
    }
}