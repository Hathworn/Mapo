#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_sub_grad(float* in1_x, float* in1_d, float* in2_x, float* in2_d, float* out, int in1ScalarCount, int in2ScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    if (tid < in1ScalarCount) {  // Ensure tid is within bounds
        int index = tid % in2ScalarCount;
        // Cooperative thread indexing for better memory usage
        for (int i = tid; i < in1ScalarCount; i += stride) {
            in1_d[i] += out[i] * in2_x[index];
            in2_d[i] = in1_x[i] * out[i];  // This is the temp array, needs to be reduced!
        }
    }
}