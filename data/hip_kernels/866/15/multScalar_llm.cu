#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void multScalar(float* in, float* out, float mult, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Minimize branch by removing condition from the loop
    while (tid < size) {
        out[tid] = in[tid] * mult;
        tid += stride;
    }
}