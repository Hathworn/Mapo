#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_log_grad(float* in_x, float* in_d, float* out_x, float* out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop for better efficiency
    for (; tid < size; tid += blockDim.x * gridDim.x) {
        if (tid < size) {
            in_d[tid] += out_d[tid] / in_x[tid];
        }
    }
}