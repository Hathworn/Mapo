#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_mul(float* in1, float* in2, float* out, int size) {
    // Compute global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop through the data in a stride pattern
    for (int i = tid; i < size; i += gridDim.x * blockDim.x) {
        // Perform element-wise multiplication
        out[i] = in1[i] * in2[i];
    }
}