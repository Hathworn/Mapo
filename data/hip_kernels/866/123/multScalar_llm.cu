#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void multScalar(float* in, float* out, float mult, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread ID is within bounds before computation
    if (tid < size) {
        out[tid] = in[tid] * mult;
    }
}