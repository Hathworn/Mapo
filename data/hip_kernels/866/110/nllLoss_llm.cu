#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nllLoss(float *x, int x_stride, float *y, int* target) {
    // Calculate the global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Assign each thread a corresponding offset
    int offset = tid * x_stride + target[tid];
    // Perform the NLL loss computation with improved readability
    y[tid] = -x[offset];
}