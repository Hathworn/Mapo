#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void arrayFill(float* data, float value, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Unrolling loop to optimize memory access and performance
    for (int i = tid; i < size; i += stride * 4) {
        data[i] = value;
        if (i + stride < size) data[i + stride] = value;
        if (i + 2 * stride < size) data[i + 2 * stride] = value;
        if (i + 3 * stride < size) data[i + 3 * stride] = value;
    }
}