#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void minusScalar(float* in, float* out, float minus, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Unroll the loop for better performance
    for (; tid < size; tid += stride * 4) {
        if (tid < size) out[tid] = in[tid] - minus;
        if (tid + stride < size) out[tid + stride] = in[tid + stride] - minus;
        if (tid + 2 * stride < size) out[tid + 2 * stride] = in[tid + 2 * stride] - minus;
        if (tid + 3 * stride < size) out[tid + 3 * stride] = in[tid + 3 * stride] - minus;
    }
}