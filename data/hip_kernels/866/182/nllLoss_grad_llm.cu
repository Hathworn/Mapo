#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nllLoss_grad(int x_stride, float *yGrad, int* target, float* xGrad) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Calculate offset conditionally to avoid out-of-bounds access
    if (tid < x_stride) {
        int offset = tid * x_stride + target[tid];
        // Utilize atomic addition to avoid race conditions if needed
        atomicAdd(&xGrad[offset], -1 * yGrad[tid]);
    }
}