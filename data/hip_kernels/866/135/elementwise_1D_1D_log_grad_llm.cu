#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_log_grad(float* in_x, float* in_d, float* out_x, float* out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Unroll loop for better performance
    while (tid < size) {
        in_d[tid] += out_d[tid] / in_x[tid];
        tid += stride;
    }
}