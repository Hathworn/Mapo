#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_minus(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use loop unrolling to enhance performance
    for (; tid < size; tid += stride * 2) {
        if (tid < size) out[tid] = in1[tid] - in2[tid];
        if (tid + stride < size) out[tid + stride] = in1[tid + stride] - in2[tid + stride];
    }
}