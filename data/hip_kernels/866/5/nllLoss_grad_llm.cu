#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void nllLoss_grad(int x_stride, float *yGrad, int* target, float* xGrad) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if tid is within bounds to prevent out-of-bounds access
    if (tid < gridDim.x * blockDim.x) {
        int offset = tid * x_stride + target[tid];
        atomicAdd(&xGrad[offset], -1.0f * yGrad[tid]); // Use atomicAdd for thread safety
    }
}