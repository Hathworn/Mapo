#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hardTanh_grad(float* in_x, float* in_d, float* out_d, float min_val, float max_val, int size, bool inplace) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    // Avoid repeated calculations by computing common comparison form once
    bool within_range;
    
    for (int i = tid; i < size; i += stride) {
        within_range = in_x[i] >= min_val && in_x[i] <= max_val;
        if (inplace) {
            // Optimize conditional check for in_d assignment
            if (!within_range) in_d[i] = 0;
        } else {
            // Use precomputed conditional result
            if (within_range) in_d[i] += out_d[i];
        }
    }
}