#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_log_grad(float* in_x, float* in_d, float* out_x, float* out_d, int size) {
    // Calculate global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use `grid-stride loop` for larger datasets
    int stride = gridDim.x * blockDim.x;

    // Unroll loop to improve performance
    for (; tid < size; tid += stride) {
        if (tid < size) {
            // Perform computation with atomic addition for accumulation
            atomicAdd(&in_d[tid], out_d[tid] / in_x[tid]);
        }
    }
}