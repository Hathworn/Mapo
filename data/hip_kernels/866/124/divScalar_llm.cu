#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void divScalar(float* in, float* out, float div, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    
    // Unrolling loop for better performance
    for (; tid + stride < size; tid += stride * 2) {
        out[tid] = in[tid] / div;
        out[tid + stride] = in[tid + stride] / div;
    }
    
    // Handle remaining elements
    if (tid < size) {
        out[tid] = in[tid] / div;
    }
}