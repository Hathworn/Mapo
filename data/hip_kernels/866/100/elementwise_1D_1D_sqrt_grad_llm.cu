#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_sqrt_grad(float* in_x, float* in_d, float* out_x, float* out_d, int size) {
    // Calculate unique thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride to cover all elements
    int stride = blockDim.x * gridDim.x;

    // Iterate over elements with stride
    while (tid < size) {
        // Efficient element-wise computation
        in_d[tid] += out_d[tid] / (out_x[tid] * 2.0f);
        tid += stride;
    }
}