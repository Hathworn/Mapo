#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask4D(float* in, int* mask, int xstrides0, int xstrides1, int xstrides2, int xstrides3, int scalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use register for linearIndex to improve performance
    for (; tid < scalarCount; tid += stride) {
        int linearIndex = tid;

        // Optimize index calculations using reduced arithmetic operations
        int xindex0 = linearIndex / xstrides0;
        linearIndex -= xstrides0 * xindex0;
        int xindex1 = linearIndex / xstrides1;
        linearIndex -= xstrides1 * xindex1;
        int xindex2 = linearIndex / xstrides2;
        int xindex3 = linearIndex - xstrides2 * xindex2;

        // Reduce memory load by caching mask[xindex0]
        int maskValue = mask[xindex0];
        if (xindex3 >= maskValue) in[tid] = 0;
    }
}