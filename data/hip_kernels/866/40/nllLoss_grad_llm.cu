#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nllLoss_grad(int x_stride, float *yGrad, int* target, float* xGrad) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < x_stride) {  // Ensure threads don't exceed array bounds
        int offset = tid * x_stride + target[tid];
        xGrad[offset] -= yGrad[tid]; // Use compound assignment for simplicity
    }
}