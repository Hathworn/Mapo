#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void nllLoss(float *x, int x_stride, float *y, int* target) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int target_value = target[tid]; // Precompute target index
    int offset = tid * x_stride + target_value; // Use precomputed target index
    y[tid] = -x[offset]; // Simplify multiplication
}