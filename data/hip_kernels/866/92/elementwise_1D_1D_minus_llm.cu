#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_minus(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Unroll loop to improve performance
    for (; tid < size; tid += stride * 4) {
        if (tid < size) out[tid] = in1[tid] - in2[tid];
        if (tid + stride < size) out[tid + stride] = in1[tid + stride] - in2[tid + stride];
        if (tid + stride * 2 < size) out[tid + stride * 2] = in1[tid + stride * 2] - in2[tid + stride * 2];
        if (tid + stride * 3 < size) out[tid + stride * 3] = in1[tid + stride * 3] - in2[tid + stride * 3];
    }
}