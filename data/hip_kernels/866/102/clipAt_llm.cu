#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void clipAt(float* in, float bound, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use loop unrolling to reduce branch divergence.
    for (; tid < size; tid += stride * 2) {
        if (tid < size) {
            float val = in[tid];
            in[tid] = (val > bound) ? bound : (val < -bound) ? -bound : val;
        }

        int next_tid = tid + stride;
        if (next_tid < size) {
            float next_val = in[next_tid];
            in[next_tid] = (next_val > bound) ? bound : (next_val < -bound) ? -bound : next_val;
        }
    }
}