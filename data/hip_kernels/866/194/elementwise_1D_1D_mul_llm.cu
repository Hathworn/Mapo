#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void elementwise_1D_1D_mul(float* in1, float* in2, float* out, int size) {
    // Calculate unique thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use stride to efficiently divide work among threads
    int stride = gridDim.x * blockDim.x;
    // Efficiently handle out-of-bound access with single check
    while (tid < size) {
        out[tid] = in1[tid] * in2[tid];
        tid += stride;
    }
}