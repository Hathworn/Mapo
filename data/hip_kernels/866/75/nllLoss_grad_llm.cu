#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nllLoss_grad(int x_stride, float *yGrad, int* target, float* xGrad) {
    // Use shared memory for efficiency if applicable
    extern __shared__ float shared_yGrad[];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tid < x_stride) {
        shared_yGrad[threadIdx.x] = yGrad[tid];
        __syncthreads(); // Ensure all threads have loaded yGrad
        
        int offset = tid * x_stride + target[tid];
        // Perform atomic operation to avoid race conditions
        atomicAdd(&xGrad[offset], -shared_yGrad[threadIdx.x]);
    }
}