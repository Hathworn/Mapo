#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hardTanh(float* in, float* out, float min_val, float max_val, int size) {
    // Calculate unique thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;

    // Unrolling the loop to process multiple elements per iteration
    for (int i = tid; i < size; i += stride * 4) {
        if (i < size) out[i] = fminf(fmaxf(in[i], min_val), max_val);
        if (i + stride < size) out[i + stride] = fminf(fmaxf(in[i + stride], min_val), max_val);
        if (i + 2 * stride < size) out[i + 2 * stride] = fminf(fmaxf(in[i + 2 * stride], min_val), max_val);
        if (i + 3 * stride < size) out[i + 3 * stride] = fminf(fmaxf(in[i + 3 * stride], min_val), max_val);
    }
}