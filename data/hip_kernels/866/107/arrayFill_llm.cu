#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void arrayFill(float* data, float value, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    // Use a loop unrolling technique for increased throughput
    for (int i = tid; i < size; i += stride * 4) {
        if (i < size) data[i] = value;
        if (i + stride < size) data[i + stride] = value;
        if (i + 2 * stride < size) data[i + 2 * stride] = value;
        if (i + 3 * stride < size) data[i + 3 * stride] = value;
    }
}