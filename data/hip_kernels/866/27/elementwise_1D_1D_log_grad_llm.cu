#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_log_grad(float* in_x, float* in_d, float* out_x, float * out_d, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Unroll loop to decrease loop overhead
    for (int i = tid; i < size; i += stride * 4) {
        if (i < size) in_d[i] += out_d[i] / in_x[i];
        if (i + stride < size) in_d[i + stride] += out_d[i + stride] / in_x[i + stride];
        if (i + 2 * stride < size) in_d[i + 2 * stride] += out_d[i + 2 * stride] / in_x[i + 2 * stride];
        if (i + 3 * stride < size) in_d[i + 3 * stride] += out_d[i + 3 * stride] / in_x[i + 3 * stride];
    }
}