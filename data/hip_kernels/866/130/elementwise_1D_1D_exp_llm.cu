#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_exp(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a conditional check to eliminate unnecessary loop iterations
    if (tid < size) {
        out[tid] = exp(in[tid]);
    }
    // Parallelize the remainder tasks using stride
    tid += blockDim.x * gridDim.x;
    while (tid < size) {
        out[tid] = exp(in[tid]);
        tid += blockDim.x * gridDim.x;
    }
}