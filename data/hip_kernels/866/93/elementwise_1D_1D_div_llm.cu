#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_div(float* in1, float* in2, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Use float comparison to avoid division by zero
    for (; tid < size; tid += stride) {
        if (tid < size) {
            float denominator = in2[tid];
            if (denominator != 0.0f) {
                out[tid] = in1[tid] / denominator;
            } else {
                out[tid] = 0.0f; // Handle division by zero gracefully
            }
        }
    }
}