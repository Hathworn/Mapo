#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void minusScalar(float* in, float* out, float minus, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    for (; tid < size; tid += stride) {
        // Combine condition with loop to prevent redundant checks
        out[tid] = in[tid] - minus;
    }
}