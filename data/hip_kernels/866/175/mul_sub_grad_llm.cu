#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_sub_grad(float* in1_x, float* in1_d, float* in2_x, float* in2_d, float* out, int in1ScalarCount, int in2ScalarCount) {
    // Calculate thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use grid stride loop for better utilization
    int stride = gridDim.x * blockDim.x;
    while (tid < in1ScalarCount) {
        int index = tid % in2ScalarCount;
        
        // Accumulate results for in1_d and in2_d
        in1_d[tid] += out[tid] * in2_x[index];
        in2_d[tid] = in1_x[tid] * out[tid];
        
        tid += stride;
    }
}