#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void momentum_update_1D_1D(float* x, float* d, float* m, float learning_rate, float momentum, float gradClip, bool nesterov, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    while (tid < size) {
        float temp = d[tid]; // Access global memory once
        if (temp > gradClip) temp = gradClip; // Gradient clipping
        if (temp < -gradClip) temp = -gradClip;
        float mt = m[tid] * momentum + temp; // Update momentum
        if (nesterov) {
            temp += momentum * mt; // Nesterov update calculation
        } else {
            temp = mt;
        }
        x[tid] -= learning_rate * temp; // Update parameter
        m[tid] = mt; // Store updated momentum
        d[tid] = 0; // Reset gradient
        tid += stride;
    }
}