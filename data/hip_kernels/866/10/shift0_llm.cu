#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void shift0(float* in, float* out, int inDim0, int inStride0, int inStride1, int inScalarCount) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Use a conditional to check array bounds first
    if(tid < inScalarCount) {
        int linearIndex = tid;
        int inIndex0 = linearIndex / inStride0;
        linearIndex = linearIndex - inIndex0 * inStride0;
        int inIndex1 = linearIndex / inStride1;
        
        // Use atomic operation for handling race conditions
        if (inIndex0 + inIndex1 < inDim0) {
            atomicExch(&out[tid + inIndex1 * inStride0], in[tid]);
        }
    }
}