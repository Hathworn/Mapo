#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multScalar(float* in, float* out, float mult, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Unrolling the loop to improve performance
    for (; tid < size; tid += stride * 4) {
        if (tid < size) out[tid] = in[tid] * mult;
        if (tid + stride < size) out[tid + stride] = in[tid + stride] * mult;
        if (tid + 2 * stride < size) out[tid + 2 * stride] = in[tid + 2 * stride] * mult;
        if (tid + 3 * stride < size) out[tid + 3 * stride] = in[tid + 3 * stride] * mult;
    }
}