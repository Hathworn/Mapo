#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum_optimization(float* in, int inStr0, int inStr1, int inStr2, int inStr3, float* out, int outStr0, int outStr1, int outStr2, int dim, int nElementOut, int dimSize) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;
    // Use float registers to reduce global memory access
    __shared__ float buffer[1024]; // Adjust size as necessary based on kernel launch parameters
    while (tid < nElementOut) { // Simplified loop iteration
        int outOff0 = tid / outStr0;
        int outOff1temp = tid - outOff0 * outStr0;
        int outOff1 = outOff1temp / outStr1;
        int outOff2 = outOff1temp - outOff1 * outStr1;
        buffer[threadIdx.x] = 0.0f; // Initialize shared memory
        for (int j = 0; j < dimSize; j++) {
            int inOff;
            if (dim == 0) inOff = j * inStr0 + outOff0 * inStr1 + outOff1 * inStr2 + outOff2 * inStr3;
            else if (dim == 1) inOff = outOff0 * inStr0 + j * inStr1 + outOff1 * inStr2 + outOff2 * inStr3;
            else if (dim == 2) inOff = outOff0 * inStr0 + outOff1 * inStr1 + j * inStr2 + outOff2 * inStr3;
            else if (dim == 3) inOff = outOff0 * inStr0 + outOff1 * inStr1 + outOff2 * inStr2 + j * inStr3;
            buffer[threadIdx.x] += in[inOff]; // Accumulate to shared memory
        }
        out[tid] = buffer[threadIdx.x]; // Write back to global memory
        tid += stride; // Move to next position
    }
}