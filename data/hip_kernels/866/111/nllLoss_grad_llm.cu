#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void nllLoss_grad(int x_stride, float *yGrad, int* target, float* xGrad) {
    // Calculate global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Check boundary condition before accessing memory
    if (tid < x_stride) {
        int offset = tid * x_stride + target[tid];
        
        // Use atomic addition for safe concurrent write
        atomicAdd(&xGrad[offset], -1 * yGrad[tid]);
    }
}