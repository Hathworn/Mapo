#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_sqrt(float* in, float* out, int size) {
    // Cache the block and thread size 
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    // Use stride and loop unrolling to reduce overhead
    for (int i = tid; i < size; i += stride) {
        out[i] = sqrt(in[i]);
    }
}