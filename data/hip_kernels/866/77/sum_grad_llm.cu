#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sum_grad(float* in, int inSize0, int inSize1, int inSize2, int inSize3, int nElement, float* out, int outStride0, int outStride1, int outStride2, int dim) {
    // Calculate thread unique index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    for (int i = tid; i < nElement; i += stride) {
        int inDim0, inDim1, inDim2, inDim3;

        // Calculate in dimension indices using efficient modulo and division
        int inOff2 = i / inSize3;
        inDim3 = i % inSize3;
        int inOff1 = inOff2 / inSize2;
        inDim2 = inOff2 % inSize2;
        inDim0 = inOff1 / inSize1;
        inDim1 = inOff1 % inSize1;

        int outOff = 0;
        // Use switch-case for better readability and possible optimizations on dim
        switch (dim) {
            case 0: outOff = inDim1 * outStride0 + inDim2 * outStride1 + inDim3 * outStride2; break;
            case 1: outOff = inDim0 * outStride0 + inDim2 * outStride1 + inDim3 * outStride2; break;
            case 2: outOff = inDim0 * outStride0 + inDim1 * outStride1 + inDim3 * outStride2; break;
            case 3: outOff = inDim0 * outStride0 + inDim1 * outStride1 + inDim2 * outStride2; break;
        }

        // Perform atomic addition to prevent race conditions
        atomicAdd(&in[i], out[outOff]);
    }
}