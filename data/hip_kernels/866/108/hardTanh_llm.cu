#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hardTanh(float* in, float* out, float min_val, float max_val, int size) {
    // Calculate the global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = gridDim.x * blockDim.x;
    
    // Use a loop with stride to handle large arrays efficiently
    for (int i = tid; i < size; i += stride) {
        // Apply min-max bound directly
        float val = in[i];
        val = val < min_val ? min_val : val;
        val = val > max_val ? max_val : val;
        out[i] = val;
    }
}