#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_div(float* in1, float* in2, float* out, int size) {
    // Use shared memory for faster access
    extern __shared__ float shared_mem[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Preload data into shared memory
    if (tid < size) {
        shared_mem[threadIdx.x] = in1[tid];
        __syncthreads();

        // Perform division using shared memory
        out[tid] = shared_mem[threadIdx.x] / in2[tid];
    }
}