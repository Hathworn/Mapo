#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_mul(float* in1, float* in2, float* out, int size) {
    // Calculate the global thread index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use 'grid-stride loop' to allow one thread to handle multiple elements
    for (int i = tid; i < size; i += blockDim.x * gridDim.x) {
        // Ensure the thread index is within bounds and perform multiplication
        if (i < size) {
            out[i] = in1[i] * in2[i];
        }
    }
}