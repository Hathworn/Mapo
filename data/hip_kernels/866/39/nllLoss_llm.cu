#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void nllLoss(float *x, int x_stride, float *y, int* target) {
    // Calculate the thread's unique global index
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate offset based on unique index and target value
    int offset = tid * x_stride + target[tid];

    // Efficient negative multiplication
    y[tid] = -x[offset];
}