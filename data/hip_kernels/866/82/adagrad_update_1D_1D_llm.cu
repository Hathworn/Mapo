#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adagrad_update_1D_1D(float* x, float* d, float* m, float clip, float lr, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop unrolling for better performance
    for (; tid < size; tid += stride) {
        // Conditionally clip the gradient
        float grad = d[tid];
        if (grad > clip) grad = clip;
        else if (grad < -clip) grad = -clip;

        // Update the mem variable and compute new parameter value
        m[tid] += grad * grad;
        x[tid] -= lr * grad / sqrt(m[tid] + 1e-8f);

        // Reset the gradient
        d[tid] = 0;
    }
}