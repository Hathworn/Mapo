#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_exp(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index
    int stride = gridDim.x * blockDim.x;  // Total number of threads in the grid
    for (; tid < size; tid += stride) {  // Loop through elements with a stride
        out[tid] = exp(in[tid]);  // Compute the exponential of input and store in output
    }
}
```
