#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void hardTanh(float* in, float* out, float min_val, float max_val, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int totalThreads = gridDim.x * blockDim.x;
    
    // Use stride loop to ensure entire input is processed
    for (int i = tid; i < size; i += totalThreads) {
        float inputVal = in[i];
        // Use fminf and fmaxf to clamp the value
        out[i] = fminf(fmaxf(inputVal, min_val), max_val);
    }
}