#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;
#ifndef MAP_FILE
#define MAP_FILE MAP_SHARED
#endif

__global__ void elementwise_1D_1D_log(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Perform computation only if tid is within range
    if (tid < size) {
        // Use a stride to allow all threads to contribute to the computation
        for (int i = tid; i < size; i += gridDim.x * blockDim.x) {
            out[i] = logf(in[i]); // Use logf for better precision and performance on floats
        }
    }
}