#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void elementwise_1D_1D_exp(float* in, float* out, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a while loop for better readability
    while (tid < size) {
        out[tid] = exp(in[tid]);
        tid += gridDim.x * blockDim.x;
    }
}