#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(double *save, double *y) {
    const int threadID = (blockIdx.x * blockDim.x + threadIdx.x) << 1;

    // Use float2 to copy two doubles at once for better memory coalescing
    reinterpret_cast<double2*>(save)[blockIdx.x * blockDim.x + threadIdx.x] = 
        reinterpret_cast<const double2*>(y)[blockIdx.x * blockDim.x + threadIdx.x];
}