#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mult3_kernel(double *g_out, double *a, double *b, double *ct, int n) {
    const int j2 = blockIdx.x * blockDim.x + threadIdx.x;
    // Use registers for frequently reused shared or calculated data
    __shared__ double shared_ct[blockDim.x];
    if (threadIdx.x < (n >> 2)) {
        shared_ct[threadIdx.x] = ct[threadIdx.x];
    }
    __syncthreads();
    
    double wkr, wki, xr, xi, yr, yi, ajr, aji, akr, aki, bjr, bji, bkr, bki;
    double new_ajr, new_aji, new_akr, new_aki;
    const int m = n >> 1;
    const int nc = n >> 2;
    const int j = j2 << 1;

    if (j2 < nc) { // Avoid accessing out-of-bounds memory
        int nminusj = n - j;

        wkr = 0.5 - shared_ct[nc - j2]; // Use shared memory for ct
        wki = shared_ct[j2]; // Use shared memory for ct

        ajr = a[j];
        aji = a[1 + j];
        akr = a[nminusj];
        aki = a[1 + nminusj];
        // Precompute repetitive operations
        xr = ajr - akr;
        xi = aji + aki;
        yr = wkr * xr - wki * xi;
        yi = wkr * xi + wki * xr;
        ajr -= yr;
        aji -= yi;
        akr += yr;
        aki -= yi;

        bjr = b[j];
        bji = b[1 + j];
        bkr = b[nminusj];
        bki = b[1 + nminusj];
        // Reuse computed wkr and wki
        xr = bjr - bkr;
        xi = bji + bki;
        yr = wkr * xr - wki * xi;
        yi = wkr * xi + wki * xr;
        bjr -= yr;
        bji -= yi;
        bkr += yr;
        bki -= yi;

        // Optimize complex multiply into fewer operations
        new_aji = ajr * bji + bjr * aji;
        new_ajr = ajr * bjr - aji * bji;
        new_aki = akr * bki + bkr * aki;
        new_akr = akr * bkr - aki * bki;

        // Reuse previous results as much as possible
        xr = new_ajr - new_akr;
        xi = new_aji + new_aki;
        yr = wkr * xr + wki * xi;
        yi = wkr * xi - wki * xr;
        g_out[j] = new_ajr - yr;
        g_out[1 + j] = yi - new_aji;
        g_out[nminusj] = new_akr + yr;
        g_out[1 + nminusj] = yi - new_aki;
    } else if (j2 == 0) {
        // Handle special case when j2 is zero separately
        xr = a[0];
        xi = a[1];
        yr = b[0];
        yi = b[1];
        g_out[0] = xr * yr + xi * yi;
        g_out[1] = -xr * yi - xi * yr;
        xr = a[0 + m];
        xi = a[1 + m];
        yr = b[0 + m];
        yi = b[1 + m];
        g_out[1 + m] = -xr * yi - xi * yr;
        g_out[0 + m] = xr * yr - xi * yi;
    }
}