#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pre_mul_kernel(int n, double *a, double *ct) {
    const int j2 = blockIdx.x * blockDim.x + threadIdx.x;
    const int nc = n >> 2;
    
    // Optimize memory access pattern
    if (j2 < nc) {
        const int j = j2 << 1;
        const int nminusj = n - j;

        // Preload values into registers
        double wkr = 0.5 - ct[nc - j2];
        double wki = ct[j2];
        double ajr = a[j];
        double aji = a[1 + j];
        double akr = a[nminusj];
        double aki = a[1 + nminusj];

        // Perform calculations
        double xr = ajr - akr;
        double xi = aji + aki;
        double yr = wkr * xr - wki * xi;
        double yi = wkr * xi + wki * xr;

        // Update results
        a[j] = ajr - yr;
        a[1 + j] = aji - yi;
        a[nminusj] = akr + yr;
        a[1 + nminusj] = aki - yi;
    }
}