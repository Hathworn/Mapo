#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sub_mul_kernel(double *g_out, double *a, double *b1, double *b2, double *ct, int n) {
    const int j2 = blockIdx.x * blockDim.x + threadIdx.x;
    const int m = n >> 1;
    const int nc = n >> 2;
    const int j = j2 << 1;
    
    // Use shared memory for ct to reduce global memory accesses
    __shared__ double shared_ct[1024]; // Assumes block size <= 1024
    if (threadIdx.x < nc) {
        shared_ct[threadIdx.x] = ct[threadIdx.x];
    }
    __syncthreads();

    if (j2) {
        int nminusj = n - j;
        double wkr = 0.5 - shared_ct[nc - j2];
        double wki = shared_ct[j2];
        double ajr = a[j];
        double aji = a[1 + j];
        double akr = a[nminusj];
        double aki = a[1 + nminusj];
        double xr = ajr - akr;
        double xi = aji + aki;
        double yr = wkr * xr - wki * xi;
        double yi = wkr * xi + wki * xr;
        ajr -= yr;
        aji -= yi;
        akr += yr;
        aki -= yi;

        // Pre-compute differences to reduce redundant calculations
        double bjr = b1[j] - b2[j];
        double bji = b1[1 + j] - b2[1 + j];
        double bkr = b1[nminusj] - b2[nminusj];
        double bki = b1[1 + nminusj] - b2[1 + nminusj];

        double new_aji = ajr * bji + bjr * aji;
        double new_ajr = ajr * bjr - aji * bji;
        double new_aki = akr * bki + bkr * aki;
        double new_akr = akr * bkr - aki * bki;

        xr = new_ajr - new_akr;
        xi = new_aji + new_aki;
        yr = wkr * xr + wki * xi;
        yi = wkr * xi - wki * xr;
        g_out[j] = new_ajr - yr;
        g_out[1 + j] = yi - new_aji;
        g_out[nminusj] = new_akr + yr;
        g_out[1 + nminusj] = yi - new_aki;
    } else {
        // Inline computation and simplify access for j2 == 0
        double xr = a[0];
        double xi = a[1];
        double yr = b1[0] - b2[0];
        double yi = b1[1] - b2[1];
        g_out[0] = xr * yr + xi * yi;
        g_out[1] = -xr * yi - xi * yr;
        xr = a[m];
        xi = a[1 + m];
        yr = b1[m] - b2[m];
        yi = b1[1 + m] - b2[1 + m];
        g_out[m] = xr * yr - xi * yi;
        g_out[1 + m] = -xr * yi - xi * yr;
    }
}