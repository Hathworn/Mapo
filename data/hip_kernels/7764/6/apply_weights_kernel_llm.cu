#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_weights_kernel(double *g_out, int *g_in, double *g_ttmp) {
    // Improved: Removed unnecessary variable declarations
    const int index = (blockIdx.x * blockDim.x + threadIdx.x) << 1;

    int val0 = g_in[index];
    int val1 = g_in[index + 1];
    double ttp_temp0 = g_ttmp[index];
    double ttp_temp1 = fabs(g_ttmp[index + 1]);

    // Simplified: Direct assignment of test without using a variable
    double coeff = ttp_temp0 < 0.0 ? -g_ttp_inc[0] : -g_ttp_inc[1];

    // Optimized: Fewer operations by combining multiplication and assignment
    g_out[index] = val0 * coeff;
    g_out[index + 1] = val1 * ttp_temp1;
}