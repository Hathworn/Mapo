#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square1_kernel(int n, double *b, double *a, double *ct) {
    const int j2 = blockIdx.x * blockDim.x + threadIdx.x;
    const int m = n >> 1;
    const int nc = n >> 2;
    const int j = j2 << 1;

    if (j2) {
        int nminusj = n - j;
        double wkr = 0.5 - ct[nc - j2];
        double wki = ct[j2];
        double ajr = a[j];
        double aji = a[1 + j];
        double akr = a[nminusj];
        double aki = a[1 + nminusj];

        // Simplified computation of new_ajr, new_aji, new_akr, new_aki.
        double new_aji = 2.0 * ajr * aji;
        double new_ajr = (ajr - aji) * (ajr + aji);
        double new_aki = 2.0 * akr * aki;
        double new_akr = (akr - aki) * (akr + aki);

        // Simplified computation for xi and xr.
        double xr = new_ajr - new_akr;
        double xi = new_aji + new_aki;

        // Using combined computation for intermediate values of yr and yi.
        double yr = wkr * xr + wki * xi;
        double yi = wkr * xi - wki * xr;

        // Directly storing results using combined computations.
        b[j] = new_ajr - yr;
        b[1 + j] = yi - new_aji;
        b[nminusj] = new_akr + yr;
        b[1 + nminusj] = yi - new_aki;
    } else {
        // Reduced redundant computations for the case when j2 equals zero.
        double xr = a[0];
        double xi = a[1];
        b[0] = xr * xr + xi * xi;
        b[1] = -xr * xi - xi * xr;
        xr = a[m];
        xi = a[1 + m];
        b[m + 1] = -xr * xi - xi * xr;
        b[m] = xr * xr - xi * xi;
    }
}