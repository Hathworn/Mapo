#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int shared_reduce(unsigned int p, volatile unsigned int * s) {
    // Optimize reduction using warp-level primitives
    s[threadIdx.x] = p;
    __syncthreads();
    
    for (unsigned int stride = warpSize / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            s[threadIdx.x] += s[threadIdx.x + stride];
        }
        __syncthreads(); // Ensure all threads have updated s before next iteration
    }
    
    return s[0]; // The reduced sum stored in s[0]
}

__global__ void reduce(unsigned int * d_out_shared, const unsigned int * d_in) {
    extern __shared__ unsigned int s[];
    int t = threadIdx.x;
    unsigned int p = d_in[t];
    unsigned int sr = shared_reduce(p, s);
    
    if (t == 0) {
        *d_out_shared = sr; // Store the result only from thread 0
    }
}