#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// http://en.wikipedia.org/wiki/Bitonic_sort

__global__ void batcherBitonicMergesort64(float *d_out, const float *d_in)
{
    // You are guaranteed this is called with <<<1, 64, 64*4>>>
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    sdata[tid] = d_in[tid];
    __syncthreads();

    // Optimize the bitonic sort stages
    for (int stage = 0; stage <= 5; stage++)
    {
        int direction = (tid >> stage) & 1;
        for (int substage = stage; substage >= 0; substage--)
        {
            int pair_tid = tid ^ (1 << substage);
            if (pair_tid > tid)
            {
                float tmp;
                if ((sdata[tid] > sdata[pair_tid]) == direction)
                {
                    tmp = sdata[tid];
                    sdata[tid] = sdata[pair_tid];
                    sdata[pair_tid] = tmp;
                }
            }
            __syncthreads();
        }
    }

    d_out[tid] = sdata[tid];
}