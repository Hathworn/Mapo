#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void global_reduce_kernel(float * d_out, float * d_in)
{
    extern __shared__ float sdata[];

    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int tid  = threadIdx.x;

    // Load shared memory
    sdata[tid] = d_in[myId];
    __syncthreads();

    // Perform reduction in shared memory for improved performance
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads(); // Ensure all threads have synchronized
    }

    // Write result back to global memory
    if (tid == 0)
    {
        d_out[blockIdx.x] = sdata[0];
    }
}