#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void optimized_histo(int *d_bins, const int *d_in, const int BIN_COUNT)
{
    int myId = threadIdx.x + blockDim.x * blockIdx.x;
    int myItem = d_in[myId];

    // Use atomic operation to prevent race condition
    atomicAdd(&d_bins[myItem % BIN_COUNT], 1);
}