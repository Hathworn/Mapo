#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void tonemap( float* d_x, float* d_y, float* d_log_Y, float* d_cdf_norm, float* d_r_new, float* d_g_new, float* d_b_new, float  min_log_Y, float  max_log_Y, float  log_Y_range, int    num_bins, int    num_pixels_y, int    num_pixels_x )
{
    int2 image_index_2d = make_int2( ( blockIdx.x * blockDim.x ) + threadIdx.x, ( blockIdx.y * blockDim.y ) + threadIdx.y );
    if ( image_index_2d.x < num_pixels_x && image_index_2d.y < num_pixels_y )
    {
        int  image_index_1d = ( num_pixels_x * image_index_2d.y ) + image_index_2d.x;
        float x = d_x[ image_index_1d ];
        float y = d_y[ image_index_1d ];
        float log_Y = d_log_Y[ image_index_1d ];
        int   bin_index = min( num_bins - 1, int( (num_bins * ( log_Y - min_log_Y ) ) / log_Y_range ) );
        float Y_new = d_cdf_norm[ bin_index ];

        float inv_y = 1.0f / y; // Precompute inverse of y for reuse
        float X_new = x * ( Y_new * inv_y );
        float Z_new = ( 1.0f - x - y ) * ( Y_new * inv_y );

        // Multiplication optimized
        float r_new = fmaf(X_new, 3.2406f, fmaf(Y_new, -1.5372f, Z_new * -0.4986f));
        float g_new = fmaf(X_new, -0.9689f, fmaf(Y_new, 1.8758f, Z_new * 0.0415f));
        float b_new = fmaf(X_new, 0.0557f, fmaf(Y_new, -0.2040f, Z_new * 1.0570f));

        d_r_new[ image_index_1d ] = r_new;
        d_g_new[ image_index_1d ] = g_new;
        d_b_new[ image_index_1d ] = b_new;
    }
}