#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void hello()
{
    // Use a single printf statement outside the loop
    if (threadIdx.x == 0) {
        for (int i = 0; i < blockDim.x; ++i) {
            printf("Hello world! I'm thread %d\n", i);
        }
    }
}