#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel function with thread and block index
__global__ void hello()
{
    // Improved inline printf statement
    printf("Hello world! I'm a thread in block %d and thread %d\n", blockIdx.x, threadIdx.x);
}