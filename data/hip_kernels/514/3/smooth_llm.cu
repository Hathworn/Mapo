#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth(float * v_new, const float * v) {
    // Compute the global index
    int myIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int numThreads = blockDim.x * gridDim.x;

    // Optimize boundary checks using min and max
    int myLeftIdx = max(0, myIdx - 1);
    int myRightIdx = min(numThreads - 1, myIdx + 1);

    // Load data
    float myElt = v[myIdx];
    float myLeftElt = v[myLeftIdx];
    float myRightElt = v[myRightIdx];

    // Compute the smoothed value
    v_new[myIdx] = 0.25f * myLeftElt + 0.5f * myElt + 0.25f * myRightElt;
}