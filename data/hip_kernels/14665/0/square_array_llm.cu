#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *a, int N)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Use built-in __powf to optimize power calculation
    if (idx < N) a[idx] = __powf(a[idx], 2.0f);
}