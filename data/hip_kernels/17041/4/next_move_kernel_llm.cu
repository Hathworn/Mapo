#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16
#define HUB_BLOCK_SIZE 32

#define TRANSITION_PROB 0.02 * 0.1

__global__ void next_move_kernel(int *rat_count, int *healthy_rat_count, int *exposed_rat_count, int *infectious_rat_count, double *node_weight, double *sum_weight_result, int *neighbor, int *neighbor_start, int width, int height, double batch_fraction) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nid = y * width + x;

    // Ensure computations are only performed within valid bounds
    if (x < width && y < height) {
        double node_weight_val = node_weight[nid];
        double sum_weight_result_val = sum_weight_result[nid];
        int rat_count_val = rat_count[nid];
        int healthy_rat_count_val = healthy_rat_count[nid];
        int exposed_rat_count_val = exposed_rat_count[nid];
        int infectious_rat_count_val = infectious_rat_count[nid];

        // Precompute move probabilities
        double batch_fraction_sum_weight = batch_fraction / sum_weight_result_val;

        for (int eid = neighbor_start[nid]; eid < neighbor_start[nid + 1]; eid++) {
            int remote_node = neighbor[eid];

            // Optimize memory access by using shared variables
            double move_prob = node_weight[remote_node] * batch_fraction_sum_weight;
            int move_rat = rat_count_val * move_prob;
            int move_healthy = healthy_rat_count_val * move_prob;
            int move_exposed = exposed_rat_count_val * move_prob;
            int move_infectious = infectious_rat_count_val * move_prob;

            // Use atomic operations to update neighbor counts
            atomicAdd(&rat_count[remote_node], move_rat);
            atomicAdd(&healthy_rat_count[remote_node], move_healthy);
            atomicAdd(&exposed_rat_count[remote_node], move_exposed);
            atomicAdd(&infectious_rat_count[remote_node], move_infectious);

            // Update current node counts
            rat_count_val -= move_rat;
            healthy_rat_count_val -= move_healthy;
            exposed_rat_count_val -= move_exposed;
            infectious_rat_count_val -= move_infectious;
        }

        // Write back the updated values
        rat_count[nid] = rat_count_val;
        healthy_rat_count[nid] = healthy_rat_count_val;
        exposed_rat_count[nid] = exposed_rat_count_val;
        infectious_rat_count[nid] = infectious_rat_count_val;
    }
}