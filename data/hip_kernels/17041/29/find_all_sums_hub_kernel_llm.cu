#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_all_sums_hub_kernel(int* hub, int nhub, double *node_weight, int *neighbor, int *neighbor_start, double *neighbor_accum_weight_result, double *sum_weight_result) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (x < nhub) {
        int nid = hub[x];
        double sum = 0.0;

        // Use shared memory to cache node weights, reducing global memory access
        extern __shared__ double shared_node_weight[];
        
        int start = neighbor_start[nid];
        int end = neighbor_start[nid + 1];
        
        for (int eid = start; eid < end; eid++) {
            // Cache node_weight in shared memory if not cached yet
            if (threadIdx.x < end - start) {
                shared_node_weight[threadIdx.x] = node_weight[neighbor[eid]];
            }
            __syncthreads();

            sum += shared_node_weight[threadIdx.x];
            neighbor_accum_weight_result[eid] = sum;

            __syncthreads();
        }

        sum_weight_result[nid] = sum;
    }
}