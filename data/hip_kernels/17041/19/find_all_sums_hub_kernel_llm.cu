#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_all_sums_hub_kernel(int* hub, int nhub, double *node_weight, int *neighbor, int *neighbor_start, double *neighbor_accum_weight_result, double *sum_weight_result) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < nhub) {
        int nid = hub[x];
        double sum = 0.0;
        int start = neighbor_start[nid];
        int end = neighbor_start[nid+1];
        // Unroll loop to improve performance and reduce loop overhead
        for (int eid = start; eid < end; eid++) {
            sum += node_weight[neighbor[eid]];
            neighbor_accum_weight_result[eid] = sum;
        }
        sum_weight_result[nid] = sum;
    }
}