#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_infection_prob_kernel(double alpha, double beta, int *infectious_rat_count, int *exposed_rat_count, int *rat_count, double *infection_prob_result, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        int nid = y * width + x;
        int current_rat_count = rat_count[nid];
        
        // Early exit if rat count is zero
        if (current_rat_count == 0) {
            infection_prob_result[nid] = 0.0;
            return;
        }
        
        double inv_rat_count = 1.0 / (double)current_rat_count; // Compute inverse once
        double density_of_exposed = (double)(exposed_rat_count[nid]) * inv_rat_count;
        double density_of_infectious = (double)(infectious_rat_count[nid]) * inv_rat_count;
        
        // Simplified calculation
        infection_prob_result[nid] = (alpha * density_of_infectious) + (beta * density_of_exposed);
    }
}