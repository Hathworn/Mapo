#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_all_sums_kernel(bool *mask, double *node_weight, int *neighbor, int *neighbor_start, double *neighbor_accum_weight_result, double *sum_weight_result, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nid = y * width + x;

    // Ensure thread is not out-of-bounds and executes only for nodes with valid mask
    if (x < width && y < height && mask[nid]) {
        double sum = 0.0;
        int start = neighbor_start[nid];
        int end = min(neighbor_start[nid + 1], start + HUB_THREASHOLD + 1); // +1 because HUB_THREASHOLD is out degree

        // Use loop unrolling to increase performance
        for (int eid = start; eid < end; eid += 4) {
            if (eid < end) sum += node_weight[neighbor[eid]];
            if (eid + 1 < end) sum += node_weight[neighbor[eid + 1]];
            if (eid + 2 < end) sum += node_weight[neighbor[eid + 2]];
            if (eid + 3 < end) sum += node_weight[neighbor[eid + 3]];

            if (eid < end) neighbor_accum_weight_result[eid] = sum;
            if (eid + 1 < end) neighbor_accum_weight_result[eid + 1] = sum;
            if (eid + 2 < end) neighbor_accum_weight_result[eid + 2] = sum;
            if (eid + 3 < end) neighbor_accum_weight_result[eid + 3] = sum;
        }

        sum_weight_result[nid] = sum;
    }
}