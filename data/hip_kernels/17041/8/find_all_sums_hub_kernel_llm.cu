#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_all_sums_hub_kernel(int* hub, int nhub, double *node_weight, int *neighbor, int *neighbor_start, double *sum_weight_result) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < nhub) {
        int nid = hub[x];
        double sum = 0.0;

        // Use shared memory for node_weight array to improve memory access efficiency
        extern __shared__ double shared_node_weight[];
        for (int i = threadIdx.x; i < nhub; i += blockDim.x) {
            shared_node_weight[i] = node_weight[i];
        }
        __syncthreads();

        // Unroll loop to reduce loop overhead
        for (int eid = neighbor_start[nid]; eid < neighbor_start[nid+1]; eid += 4) {
            sum += shared_node_weight[neighbor[eid]];
            if (eid+1 < neighbor_start[nid+1]) sum += shared_node_weight[neighbor[eid+1]];
            if (eid+2 < neighbor_start[nid+1]) sum += shared_node_weight[neighbor[eid+2]];
            if (eid+3 < neighbor_start[nid+1]) sum += shared_node_weight[neighbor[eid+3]];
        }
        sum_weight_result[nid] = sum;
    }
}