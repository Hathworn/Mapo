#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void next_move_hub_kernel(int* hub, int nhub, int *rat_count, int *healthy_rat_count, int *exposed_rat_count, int *infectious_rat_count, double *node_weight, double *sum_weight_result, int *neighbor, int *neighbor_start, int width, int height, double batch_fraction) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize: Use shared memory to reduce global memory access latency
    __shared__ double shared_node_weight[MAX_THREADS]; // Adjust size according to the maximum number of threads
    __shared__ double shared_sum_weight_result[MAX_THREADS];

    if (x < nhub) {
        int nid = hub[x];
        shared_node_weight[threadIdx.x] = node_weight[nid];
        shared_sum_weight_result[threadIdx.x] = sum_weight_result[nid];
        __syncthreads();
        
        for (int eid = neighbor_start[nid]; eid < neighbor_start[nid+1]; eid++) {
            int remote_node = neighbor[eid];
            
            // Use shared memory values
            double move_prob = batch_fraction * shared_node_weight[threadIdx.x] / shared_sum_weight_result[threadIdx.x];
            
            int move_rat = rat_count[nid] * move_prob;
            int move_healthy = healthy_rat_count[nid] * move_prob;
            int move_exposed = exposed_rat_count[nid] * move_prob;
            int move_infectious = infectious_rat_count[nid] * move_prob;
            
            atomicAdd(&rat_count[remote_node], move_rat);
            atomicAdd(&healthy_rat_count[remote_node], move_healthy);
            atomicAdd(&exposed_rat_count[remote_node], move_exposed);
            atomicAdd(&infectious_rat_count[remote_node], move_infectious);

            rat_count[nid] -= move_rat;
            healthy_rat_count[nid] -= move_healthy;
            exposed_rat_count[nid] -= move_exposed;
            infectious_rat_count[nid] -= move_infectious;
        }
    }
}