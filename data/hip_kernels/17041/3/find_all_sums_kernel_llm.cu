#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16
#define HUB_BLOCK_SIZE 32
#define TRANSITION_PROB 0.02 * 0.1

__global__ void find_all_sums_kernel(bool *mask, double *node_weight, int *neighbor, int *neighbor_start, double *sum_weight_result, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nid = y * width + x; // thread_index is node id

    // Check if the current thread is within bounds
    if (x < width && y < height && mask[nid]) {
        double sum = 0.0;
        
        // Prefetch neighbor start indices
        int start = neighbor_start[nid];
        int end = neighbor_start[nid + 1];

        // Clamp end for threshold and loop improvement
        int clamped_end = min(end, start + HUB_THREASHOLD + 1);
        
        for (int eid = start; eid < clamped_end; eid++) {
            // Accumulate weights of neighbors
            sum += node_weight[neighbor[eid]];
        }
        
        // Store computed sum in result array
        sum_weight_result[nid] = sum;
    }
}