#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16
#define HUB_BLOCK_SIZE 32

#define TRANSITION_PROB 0.02 * 0.1

__global__ void find_all_sums_hub_kernel(int* hub, int nhub, double *node_weight, int *neighbor, int *neighbor_start, double *sum_weight_result) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < nhub) {
        int nid = hub[x];
        double sum = 0.0;
        
        // Use of register for frequently used neighbor_start[nid]
        int start = neighbor_start[nid];
        int end = neighbor_start[nid+1];
        
        // Using unrolling for loop to improve performance
        int eid;
        for (eid = start; eid < end - 4; eid += 4) {
            sum += node_weight[neighbor[eid]];
            sum += node_weight[neighbor[eid + 1]];
            sum += node_weight[neighbor[eid + 2]];
            sum += node_weight[neighbor[eid + 3]];
        }
        
        // Handle leftover iterations
        for (; eid < end; eid++) {
            sum += node_weight[neighbor[eid]];
        }
        
        sum_weight_result[nid] = sum;
    }
}