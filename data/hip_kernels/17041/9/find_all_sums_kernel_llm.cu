#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_all_sums_kernel(bool *mask, double *node_weight, int *neighbor, int *neighbor_start, double *sum_weight_result, int width, int height) {
    // Calculate unique thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nid = y * width + x;
    
    // Early exit if thread is out of bounds or mask is false
    if (x >= width || y >= height || !mask[nid]) return;

    double sum = 0.0;
    
    // Use shared memory to improve performance by reducing global memory access
    extern __shared__ double shared_weights[];
    int end = min(neighbor_start[nid + 1], neighbor_start[nid] + HUB_THREASHOLD + 1);
    
    // Load weights into shared memory
    for (int eid = neighbor_start[nid] + threadIdx.x; eid < end; eid += blockDim.x) {
        shared_weights[threadIdx.x] = node_weight[neighbor[eid]];
        __syncthreads();
        
        // Perform reduction in shared memory
        for (int s = blockDim.x / 2; s > 0; s >>= 1) {
            if (threadIdx.x < s) {
                shared_weights[threadIdx.x] += shared_weights[threadIdx.x + s];
            }
            __syncthreads();
        }
        
        // Only thread 0 writes the result
        if (threadIdx.x == 0) {
            sum += shared_weights[0];
        }
    }
    
    sum_weight_result[nid] = sum;
}