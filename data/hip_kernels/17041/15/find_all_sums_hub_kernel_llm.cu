#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_all_sums_hub_kernel(int* hub, int nhub, double *node_weight, int *neighbor, int *neighbor_start, double *neighbor_accum_weight_result, double *sum_weight_result) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure we're within bounds and load hub index
    if (x < nhub) {
        int nid = hub[x];
        double sum = 0.0;
        
        // Use shared memory for thread-local sums optimization
        extern __shared__ double shared_sum[];
        shared_sum[threadIdx.x] = 0.0;

        // Prefetch neighbor_start values for coalesced global memory access
        int start = neighbor_start[nid];
        int end = neighbor_start[nid + 1];

        // Accumulate weights in local shared memory
        for (int eid = start; eid < end; eid++) {
            shared_sum[threadIdx.x] += node_weight[neighbor[eid]];
            neighbor_accum_weight_result[eid] = shared_sum[threadIdx.x];
        }
        
        // Write the final summed value to global memory
        sum_weight_result[nid] = shared_sum[threadIdx.x];
    }
}