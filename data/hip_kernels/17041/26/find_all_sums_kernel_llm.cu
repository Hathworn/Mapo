#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_all_sums_kernel(bool *mask, float *node_weight, int *neighbor, int *neighbor_start, float *neighbor_accum_weight_result, float *sum_weight_result, int width, int height){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nid = y * width + x; // thread_index is node id

    // Check for valid thread to process
    if (x < width && y < height && mask[nid]){
        float sum = 0.0;
        
        // Use shared memory for accumulative sum (assuming a block has enough resources)
        __shared__ float shared_weight[BLOCK_SIZE];
        
        // Efficient memory access
        int start = neighbor_start[nid];
        int end = min(neighbor_start[nid + 1], start + HUB_THREASHOLD + 1);
        
        // Load data into shared memory, each thread handles one neighbor
        for (int eid = start + threadIdx.x; eid < end; eid += blockDim.x) {
            shared_weight[threadIdx.x] = node_weight[neighbor[eid]];
            __syncthreads();

            // Accumulate sum
            if (threadIdx.x == 0) {
                for (int i = start; i < end; i++) {
                    sum += shared_weight[i - start];
                    neighbor_accum_weight_result[i] = sum;
                }
            }

            // Ensure all threads have updated the weight before next iteration
            __syncthreads();
        }
        
        if (threadIdx.x == 0) {
            sum_weight_result[nid] = sum;
        }
    }
}