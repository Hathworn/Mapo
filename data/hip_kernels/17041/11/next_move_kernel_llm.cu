#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void next_move_kernel(bool *mask, int *rat_count, int *healthy_rat_count, int *exposed_rat_count, int *infectious_rat_count, double *node_weight, double *sum_weight_result, int *neighbor, int *neighbor_start, int width, int height, double batch_fraction) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nid = y * width + x;

    if (x < width && y < height) {
        int start = neighbor_start[nid];
        int end = min(start + 5, neighbor_start[nid + 1]); // Limit to valid range

        double local_sum_weight_result = sum_weight_result[nid];
        int local_rat_count = rat_count[nid];
        int local_healthy_rat_count = healthy_rat_count[nid];
        int local_exposed_rat_count = exposed_rat_count[nid];
        int local_infectious_rat_count = infectious_rat_count[nid];

        // Loop unrolling for efficiency
        for (int eid = start; eid < end; eid++) {
            int remote_node = neighbor[eid];
            double node_weight_remote = node_weight[remote_node];
            double move_prob = batch_fraction * node_weight_remote / local_sum_weight_result;

            int move_rat = __double2int_rn(local_rat_count * move_prob);
            int move_healthy = __double2int_rn(local_healthy_rat_count * move_prob);
            int move_exposed = __double2int_rn(local_exposed_rat_count * move_prob);
            int move_infectious = __double2int_rn(local_infectious_rat_count * move_prob);

            atomicAdd(&rat_count[remote_node], move_rat);
            atomicAdd(&healthy_rat_count[remote_node], move_healthy);
            atomicAdd(&exposed_rat_count[remote_node], move_exposed);
            atomicAdd(&infectious_rat_count[remote_node], move_infectious);

            local_rat_count -= move_rat;
            local_healthy_rat_count -= move_healthy;
            local_exposed_rat_count -= move_exposed;
            local_infectious_rat_count -= move_infectious;
        }

        // Store back to global memory
        rat_count[nid] = local_rat_count;
        healthy_rat_count[nid] = local_healthy_rat_count;
        exposed_rat_count[nid] = local_exposed_rat_count;
        infectious_rat_count[nid] = local_infectious_rat_count;
    }
}