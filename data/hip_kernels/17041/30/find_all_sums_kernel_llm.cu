#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_all_sums_kernel(bool *mask, double *node_weight, int *neighbor, int *neighbor_start, double *neighbor_accum_weight_result, double *sum_weight_result, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nid = y * width + x; // thread_index is node id

    // Check if x and y are within bounds and if mask[nid] is true
    if (x < width && y < height && mask[nid]) {
        double sum = 0.0;
        int end = min(neighbor_start[nid+1], neighbor_start[nid]+HUB_THREASHOLD+1); // Limit loop upper bound

        // Unroll loop for better performance if the number of iterations is small
        #pragma unroll 
        for (int eid = neighbor_start[nid]; eid < end; eid++) { 
            sum += node_weight[neighbor[eid]];
            neighbor_accum_weight_result[eid] = sum;
        }
        sum_weight_result[nid] = sum;
    }
}