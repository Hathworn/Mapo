#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_all_sums_kernel(bool *mask, double *node_weight, int *neighbor, int *neighbor_start, double *neighbor_accum_weight_result, double *sum_weight_result, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int nid = y * width + x; // thread_index is node id

    // Ensure the node is within valid bounds and is active
    if (x < width && y < height && mask[nid]) {
        double sum = 0.0;
        int start = neighbor_start[nid];
        int end = min(neighbor_start[nid + 1], start + HUB_THREASHOLD + 1); // +1 because HUB_THREADHOLD is out degree

        // Accumulate neighbor weights
        for (int eid = start; eid < end; eid++) {
            sum += node_weight[neighbor[eid]];
            neighbor_accum_weight_result[eid] = sum;
        }
        sum_weight_result[nid] = sum;
    }
}