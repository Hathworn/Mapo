#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void find_all_sums_hub_kernel(int* hub, int nhub, float *node_weight, int *neighbor, int *neighbor_start, float *neighbor_accum_weight_result, float *sum_weight_result) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < nhub) {
        int nid = hub[x];
        float sum = 0.0;

        // Use registers to minimize global memory access and avoid repeated computations
        int start = neighbor_start[nid];
        int end = neighbor_start[nid + 1];

        for (int eid = start; eid < end; eid++) {
            sum += node_weight[neighbor[eid]];
            neighbor_accum_weight_result[eid] = sum;
        }
        sum_weight_result[nid] = sum;
    }
}