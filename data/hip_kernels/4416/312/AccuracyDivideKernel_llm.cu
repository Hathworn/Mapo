#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {  // Ensure only one thread performs the division
        *accuracy /= N;
    }
}