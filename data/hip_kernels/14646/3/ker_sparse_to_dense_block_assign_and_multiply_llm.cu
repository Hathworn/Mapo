#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_sparse_to_dense_block_assign_and_multiply(int n, const unsigned *idx, int bsize, float mult, float* src, float *trg) {
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the source index
    int src_index = idx[id / bsize] * bsize + id % bsize;

    // Make sure we do not go out of bounds
    if (id < n * bsize) {
        // Perform multiplication and assignment
        trg[id] = src[src_index] * mult;
    }
}