#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void ker_dense_to_sparse_block_add(int n, const unsigned *idx, int bsize, float* src, float *trg) {
    // Use shared memory for intermediate storage to reduce global memory accesses
    __shared__ float sharedSrc[256]; // Adjust size as needed

    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int localId = threadIdx.x;

    // Make sure we do not go out of bounds
    if (id < n * bsize) {
        // Copy to shared memory
        sharedSrc[localId] = src[id];
        __syncthreads();

        // Perform atomic addition using shared memory value
        atomicAdd(trg + idx[id / bsize] * bsize + id % bsize, sharedSrc[localId]);
    }
}