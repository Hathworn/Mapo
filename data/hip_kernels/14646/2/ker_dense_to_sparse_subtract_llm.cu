#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Improved kernel function with coalesced memory access
__global__ void ker_dense_to_sparse_subtract(int n, const unsigned int *idx, float *src, float *trg) {
    // Get our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Coalesced memory access and prevent out-of-bounds access
    if (id < n) {
        unsigned int target_idx = idx[id];
        float src_val = src[id];
        atomicAdd(&trg[target_idx], -src_val);
    }
}