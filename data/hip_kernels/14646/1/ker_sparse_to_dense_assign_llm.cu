#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ker_sparse_to_dense_assign(int n, const unsigned int *idx, float *src, float *trg) {
    // Calculate our global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure we are within bounds before accessing memory
    if (id < n) {
        // Fetch and assign without redundant memory access
        unsigned int index = idx[id];
        trg[id] = src[index];
    }
}