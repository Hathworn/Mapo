#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AddIntsCUDA(int* a, int* b) {
    // Use shared memory for faster access
    __shared__ int temp_a;
    __shared__ int temp_b;
    
    // Load data into shared memory
    if (threadIdx.x == 0) {
        temp_a = a[0];
        temp_b = b[0];
    }
    __syncthreads();

    // Use a single thread to perform addition and avoid redundant global memory access
    if (threadIdx.x == 0) {
        for (int i = 0; i < 1000005; i++) {
            temp_a += temp_b;
        }
    }
    __syncthreads();

    // Write result back to global memory
    if (threadIdx.x == 0) {
        a[0] = temp_a;
    }
}