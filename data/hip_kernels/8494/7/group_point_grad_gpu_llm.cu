#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_grad_gpu(int b, int n, int c, int m, int nsample, const float *grad_out, const int *idx, float *grad_points) {
    int batch_index = blockIdx.x;
    idx += m * nsample * batch_index;
    grad_out += m * nsample * c * batch_index;
    grad_points += n * c * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;
    
    // Optimize loop to minimize thread divergence
    for (int j=index; j<m; j+=stride) {
        int base_idx = j * nsample;
        for (int k=0; k<nsample; ++k) {
            int ii = idx[base_idx + k];
            int base_grad = base_idx * c + k * c;
            for (int l=0; l<c; ++l) {
                atomicAdd(&grad_points[ii * c + l], grad_out[base_grad + l]);
            }
        }
    }
}