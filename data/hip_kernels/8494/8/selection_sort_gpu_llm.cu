#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void selection_sort_gpu(int b, int n, int m, int k, const float *dist, int *outi, float *out) {
    int batch_index = blockIdx.x;
    dist += m * n * batch_index;
    outi += m * n * batch_index;
    out += m * n * batch_index;
    
    int index = threadIdx.x;
    int stride = blockDim.x;
    
    // Optimize: Unroll loop for copy operation
    for (int j = index; j < m; j += stride) {
        #pragma unroll
        for (int s = 0; s < n; ++s) {
            out[j * n + s] = dist[j * n + s];
            outi[j * n + s] = s;
        }
    }
    
    float *p_dist;
    for (int j = index; j < m; j += stride) {
        p_dist = out + j * n;
        
        // Optimize: Unroll selection sort for the first k elements
        for (int s = 0; s < k; ++s) {
            int min = s;
            
            for (int t = s + 1; t < n; ++t) {
                if (p_dist[t] < p_dist[min]) {
                    min = t;
                }
            }
            
            if (min != s) {
                // Optimize: Avoid temporary variables by using tuple swap
                swap(p_dist[min], p_dist[s]);
                swap(outi[j * n + min], outi[j * n + s]);
            }
        }
    }
}