#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void group_point_gpu(int b, int n, int c, int m, int nsample, const float *points, const int *idx, float *out) {
    int batch_index = blockIdx.x;
    points += n * c * batch_index;
    idx += m * nsample * batch_index;
    out += m * nsample * c * batch_index;

    int index = threadIdx.x;
    int stride = blockDim.x;

    // Use shared memory to reduce global memory access for points
    __shared__ float shared_points[1024]; // Adjust size as needed
    for (int j = index; j < m; j += stride) {
        for (int k = 0; k < nsample; ++k) {
            int ii = idx[j * nsample + k];
            // Load points into shared memory
            if (index < c) {
                shared_points[index] = points[ii * c + index];
            }
            __syncthreads();
            for (int l = 0; l < c; ++l) {
                out[j * nsample * c + k * c + l] = shared_points[l];
            }
            __syncthreads();
        }
    }
}