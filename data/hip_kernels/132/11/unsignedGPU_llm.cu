#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void unsignedGPU(int numTests, unsigned* ns, unsigned* ds, unsigned* qs, unsigned* rs) {
    // Calculate global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if thread index is within bounds
    if (idx < numTests) {
        unsigned n = ns[idx];
        unsigned d = ds[idx];

        qs[idx] = n / d;  // Compute quotient
        rs[idx] = n % d;  // Compute remainder
    }
}