#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void signedGPU(int numTests, int* ns, int* ds, int* qs, int* rs) {
    // Parallelize computation using block and thread indices
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if idx is within bounds to avoid out-of-bounds access
    if (idx < numTests) {
        int n = ns[idx];
        int d = ds[idx];

        qs[idx] = n / d;
        rs[idx] = n % d;
    }
}