#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sleepKernel(double* cycles, int64_t waitCycles) {
    extern __shared__ int s[];
    long long int start = clock64();

    // Unroll the loop to decrease the number of branching
    while (true) {
        // Cache the clock64() value to reduce register pressure
        long long int current = clock64();
        long long int total = current - start;

        if (total >= waitCycles) {
            break;
        }
    }
    *cycles = static_cast<double>(clock64() - start);
}