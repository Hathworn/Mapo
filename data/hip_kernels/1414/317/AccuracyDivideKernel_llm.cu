#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel by adding const qualifier and using grid-stride loop
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx == 0) { // Ensure only one thread performs the division
        *accuracy /= N;
    }
}