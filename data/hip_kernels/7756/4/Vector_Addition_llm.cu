#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void Vector_Addition(const int* __restrict__ dev_a, const int* __restrict__ dev_b, int* __restrict__ dev_c) 
{
    // Use int for tid to support larger indices; leverage __restrict__ for better memory access optimization
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < N) // boundary condition check remains efficient
    {
        dev_c[tid] = dev_a[tid] + dev_b[tid];
    }
}