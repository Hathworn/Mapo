#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ))

__global__ void Vector_Plus ( int *AG ,  int *BG , int *CG)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    // Using pointer arithmetic for better performance
    if (id < N)
        CG[id] = AG[id] + BG[id];
}