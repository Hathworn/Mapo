#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixAdd_CUDA(int *A, int *B, int *C, int width) {
    // Calculate the row index of the C matrix
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    // Calculate the column index of the C matrix
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    // Check bounds before accessing memory
    if (i < width && j < width) {
        // Perform matrix addition for the current element
        C[i * width + j] = A[i * width + j] + B[i * width + j];
    }
}