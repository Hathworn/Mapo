#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void Vector_Addition(int *dev_a, int *dev_b, int *dev_c)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if thread ID is within bounds and perform addition
    if (tid < N)
        dev_c[tid] = dev_a[tid] + dev_b[tid];
}