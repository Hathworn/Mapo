#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Macros
#define min(a, b) ( (a)<(b)? (a): (b) )
#define max(a, b) ( (a)>(b)? (a): (b) )

// Constants
#define MAX_VECTOR_COUNT 5

// Vector structure
typedef struct {
    float e[3];
} Vec3f;

// Global array
Vec3f vecArray[MAX_VECTOR_COUNT];
Vec3f newvecArray[MAX_VECTOR_COUNT];

// Forward declarations

__global__ void reduce(Vec3f *input, Vec3f *output) {
    extern __shared__ Vec3f sdata[];

    // Each thread loadsome element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
    sdata[tid] = input[i];
    __syncthreads();

    // Perform reduction in shared mem
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {  // Use bitwise shift and reverse loop direction
        if (tid < s) {
            // Summing
            sdata[tid].e[0] += sdata[tid + s].e[0];
            sdata[tid].e[1] += sdata[tid + s].e[1];
            sdata[tid].e[2] += sdata[tid + s].e[2];
        }
        __syncthreads();
    }

    // Write result for this block to global mem
    if (tid == 0) output[blockIdx.x] = sdata[0];
}