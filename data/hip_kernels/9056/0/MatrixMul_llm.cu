#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define THREADS_PER_BLOCK 256

__global__ void MatrixMul(float *Md, float *Nd, float *Pd, const int WIDTH) {
    // Compute column and row for the element
    int COL = threadIdx.x + blockIdx.x * blockDim.x;
    int ROW = threadIdx.y + blockIdx.y * blockDim.y;

    if (ROW < WIDTH && COL < WIDTH) {
        float Pvalue = 0; // Accumulator for the current element
        // Use shared memory for partial submatrices
        __shared__ float Mds[THREADS_PER_BLOCK][THREADS_PER_BLOCK];
        __shared__ float Nds[THREADS_PER_BLOCK][THREADS_PER_BLOCK];

        // Loop over phases
        for (int ph = 0; ph < WIDTH / THREADS_PER_BLOCK; ++ph) {
            // Load Md and Nd tiles into shared memory
            Mds[threadIdx.y][threadIdx.x] = Md[ROW * WIDTH + (ph * THREADS_PER_BLOCK + threadIdx.x)];
            Nds[threadIdx.y][threadIdx.x] = Nd[(ph * THREADS_PER_BLOCK + threadIdx.y) * WIDTH + COL];
            __syncthreads();

            // Multiply tiles
            for (int k = 0; k < THREADS_PER_BLOCK; ++k) {
                Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
            }
            __syncthreads();
        }
        // Write back the result
        Pd[ROW * WIDTH + COL] = Pvalue;
    }
}