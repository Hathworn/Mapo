#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMul(float *Md, float *Nd, float *Pd, const int WIDTH)
{
    // Calculate global thread row and column indices
    int COL = threadIdx.x + blockIdx.x * blockDim.x;
    int ROW = threadIdx.y + blockIdx.y * blockDim.y;

    // Initialize the sum to zero
    float sum = 0.0f;

    // Check if within bounds
    if (ROW < WIDTH && COL < WIDTH) {
        // Perform the dot product for a single element
        for (int i = 0; i < WIDTH; i++) {
            sum += Md[ROW * WIDTH + i] * Nd[i * WIDTH + COL];
        }
        Pd[ROW * WIDTH + COL] = sum; // Store the result
    }
}