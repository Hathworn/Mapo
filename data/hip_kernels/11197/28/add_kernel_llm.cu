#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the global index more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use strided access to cover all elements
    int stride = blockDim.x * gridDim.x;
    for (; i < N; i += stride) {
        X[i * INCX] += ALPHA;
    }
}