#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Optimize bounds check
    if (i >= n) return;

    // Cache frequently used values and minimize recomputations
    float diff = truth[i] - pred[i];
    float abs_val = fabsf(diff);  // Use `fabsf` for single-precision

    // Simplify branching logic for performance
    if (abs_val < 1) {
        error[i] = diff * diff;
        delta[i] = diff;
    } else {
        error[i] = abs_val * 2.0f - 1.0f;  // Use floating-point literals for consistency
        delta[i] = copysignf(1.0f, diff);   // Use `copysignf` for clarity and optimization
    }
}