#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    // Precompute constant for variance scaling
    float variance_scale = -.5f * powf(variance[i] + .00001f, -1.5f);
    float sum = 0.0f;

    // Unrolling loop for better performance
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += delta[index] * (x[index] - mean[i]);
        }
    }

    variance_delta[i] = sum * variance_scale; // Apply scaling after accumulation
}