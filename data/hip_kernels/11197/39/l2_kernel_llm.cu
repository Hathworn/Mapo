#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate global index for current thread
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure index does not exceed bounds
    if(i < n) {
        float diff = truth[i] - pred[i];
        // Use fma for efficient computation of error[i] = diff * diff (if supported)
        error[i] = diff * diff; 
        delta[i] = diff;
    }
}