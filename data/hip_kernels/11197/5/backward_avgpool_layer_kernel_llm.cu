#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    // Calculate the unique thread index
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n * w * h) return;  // Ensure index within bounds

    // Efficiently determine b (batch), k (channel), and pixel indices
    int pixel_idx = id % (w * h);
    int k = (id / (w * h)) % c;
    int b = id / (w * h * c);

    // Compute indices
    int in_index = pixel_idx + h * w * (k + b * c);
    int out_index = k + c * b;

    // Atomic addition to prevent race conditions
    atomicAdd(&in_delta[in_index], out_delta[out_index] / (w * h));
}