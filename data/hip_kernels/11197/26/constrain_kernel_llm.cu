#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure i increments correctly by INCX
    i *= INCX;

    // Use grid-stride loop to allow more threads to contribute
    for (; i < N; i += blockDim.x * gridDim.x) {
        // Clamp X[i] to the range [-ALPHA, ALPHA]
        X[i] = fminf(ALPHA, fmaxf(-ALPHA, X[i]));
    }
}