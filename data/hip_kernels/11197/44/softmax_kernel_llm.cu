#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    int i;
    float sum = 0;
    float largest = -INFINITY;
    for(i = 0; i < n; ++i){
        float val = input[i*stride];  // Use float for val instead of int
        largest = (val>largest) ? val : largest;
    }
    for(i = 0; i < n; ++i){
        float e = expf((input[i*stride] - largest) / temp);  // Optimize by combining expressions and using expf
        sum += e;
        output[i*stride] = e;
    }
    float inv_sum = 1.0f / sum;  // Compute inverse of sum once
    for(i = 0; i < n; ++i){
        output[i*stride] *= inv_sum;  // Multiply by inverse to divide
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;  // Optimize index calculation
    if (id < batch * groups) {  // Simplify boundary check
        int b = id / groups;
        int g = id % groups;
        softmax_device(input + b * batch_offset + g * group_offset, n, temp, stride, output + b * batch_offset + g * group_offset);
    }
}