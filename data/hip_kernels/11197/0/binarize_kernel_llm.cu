#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation for 1D grid
    if (i < n) {  // Changed comparison to process within bounds
        binary[i] = (x[i] >= 0) ? 1 : -1;  // Perform binary operation
    }
}