#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f;

    // Optimize loop by unrolling
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index] * x_norm[index];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Use reduction pattern for sum
    if (p < 32) {
        part[p] += part[p + 32];
        part[p] += part[p + 16];
        part[p] += part[p + 8];
        part[p] += part[p + 4];
        part[p] += part[p + 2];
        part[p] += part[p + 1];
    }
    
    if (p == 0) {
        scale_updates[filter] += part[0];
    }
}