#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= groups) return;
    
    float local_sum = 0.0f; // Use local variable for sum to reduce global memory transactions
    for (int k = 0; k < n; ++k) {
        local_sum += x[k * groups + i]; // Accumulate directly into local_sum
    }

    sum[i] = local_sum; // Assign final result to global memory once
}