#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a stride loop for larger arrays
    for(int i = id; i < size; i += blockDim.x * gridDim.x) {
        // Conditional operation on input
        input[i] = (rand[i] < prob) ? 0 : input[i] * scale;
    }
}