#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;  // Calculate stride for grid-stride loop
    while (index < N) {
        int f = (index/spatial) % filters;
        x[index] = (x[index] - mean[f]) / (sqrtf(variance[f] + .00001f));
        index += stride;  // Increment index by stride
    }
}