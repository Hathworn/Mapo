#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, int w1, int h1, int c1, float *add, int w2, int h2, int c2, float *out)
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if thread ID is within size
    if (id >= size) return;
    
    // Precompute multipliers
    int minw_minh = minw * minh;
    int minw_minh_minc = minw_minh * minc;

    // Calculate indices
    int b = id / minw_minh_minc;
    int remain = id % minw_minh_minc;
    int k = remain / minw_minh;
    remain = remain % minw_minh;
    int j = remain / minw;
    int i = remain % minw;
    
    // Calculate indices for output and addition arrays
    int out_index = i * sample + w2 * (j * sample + h2 * (k + c2 * b));
    int add_index = i * stride + w1 * (j * stride + h1 * (k + c1 * b));
    
    // Perform addition
    out[out_index] += add[add_index];
}