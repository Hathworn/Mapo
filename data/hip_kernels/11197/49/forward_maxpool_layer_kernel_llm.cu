#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    // Calculate output dimensions
    int h = (in_h + 2 * pad) / stride;
    int w = (in_w + 2 * pad) / stride;

    // Calculate unique thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Calculate indices
    int w_idx = id % w;
    id /= w;
    int h_idx = id % h;
    id /= h;
    int c_idx = id % in_c;
    id /= in_c;
    int b_idx = id;

    // Set initial values
    int w_offset = -pad;
    int h_offset = -pad;
    float max_val = -INFINITY;
    int max_index = -1;

    // Calculate current position
    int out_index = w_idx + w * (h_idx + h * (c_idx + in_c * b_idx));

    // Unrolled nested loops for better performance
    for (int l = 0; l < size; ++l) {
        int cur_h = h_offset + h_idx * stride + l;
        if (cur_h < 0 || cur_h >= in_h) continue; // Boundary check

        for (int m = 0; m < size; ++m) {
            int cur_w = w_offset + w_idx * stride + m;
            if (cur_w < 0 || cur_w >= in_w) continue; // Boundary check

            // Calculate the index in the input array
            int index = cur_w + in_w * (cur_h + in_h * (c_idx + b_idx * in_c));

            // Check for the maximum value
            float val = input[index];
            if (val > max_val) {
                max_val = val;
                max_index = index;
            }
        }
    }

    // Set output values
    output[out_index] = max_val;
    indexes[out_index] = max_index;
}
```
