#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    // Calculate the global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index >= N) return;

    // Calculate indices based on flattened index
    int b = index / (w * h * c);
    int in_c = (index / (w * h)) % c;
    int in_h = (index / w) % h;
    int in_w = index % w;

    // Calculate the output coordinates
    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    int out_index = b * out_c * h * stride * w * stride + c2 * h * stride * w * stride + h2 * w * stride + w2;

    // Perform the reorganization
    if(forward) 
        out[out_index] = x[index];
    else 
        out[index] = x[out_index];
}