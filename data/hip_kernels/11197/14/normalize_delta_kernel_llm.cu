#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    // Optimized block and grid index computation
    int index = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    if (index >= N) return;

    int f = (index / spatial) % filters;
    float var_sqrt_inv = rsqrtf(variance[f] + 0.00001f); // Precompute inverse sqrt for normalization
    float scale = 1.0f / (spatial * batch);

    // Use precomputed values and simplified operations
    delta[index] = delta[index] * var_sqrt_inv 
                 + variance_delta[f] * 2.0f * (x[index] - mean[f]) * scale 
                 + mean_delta[f] * scale;
}