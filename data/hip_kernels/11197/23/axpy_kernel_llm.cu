#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX,  float *Y, int OFFY, int INCY)
{
    // Calculate the unique thread index across the grid 
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a grid-stride loop for better memory access and to handle any extra threads
    for (; i < N; i += blockDim.x * gridDim.x) {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}