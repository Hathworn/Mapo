#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = blockIdx.x * blockDim.x + threadIdx.x; // Simplified block indexing
    if (f >= n) return;
    
    float mean = 0.0f;
    #pragma unroll // Encourage loop unrolling for increased performance
    for(int i = 0; i < size; ++i){
        mean += abs(weights[f * size + i]);
    }
    mean = mean / size;

    #pragma unroll // Encourage loop unrolling for increased performance
    for(int i = 0; i < size; ++i){
        binary[f * size + i] = (weights[f * size + i] > 0) ? mean : -mean;
    }
}