#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    // Use a more efficient loop and load balancing by unrolling the loop
    for(int j = 0; j < batch; ++j){
        for(int i = id; i < spatial; i += threads){
            int index = j * spatial * filters + filter * spatial + i;
            if(i < spatial) local[id] += delta[index];
        }
    }

    __syncthreads();

    // Use atomic operation to prevent race conditions when combining results
    if(id == 0){
        float sum = 0.0f;
        for(int i = 0; i < threads; ++i){
            sum += local[i];
        }
        mean_delta[filter] = sum * (-1.f / sqrtf(variance[filter] + .00001f));
    }
}