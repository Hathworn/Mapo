#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate unique thread index within grid
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop to allow one thread to process multiple elements
    int stride = gridDim.x * blockDim.x;
    for (; i < n; i += stride) {
        c[i] += a[i] * b[i];
    }
}