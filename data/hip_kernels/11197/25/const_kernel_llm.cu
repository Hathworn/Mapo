#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Use efficient loop for processing in case N > total threads
    while (i < N) {
        X[i * INCX] = ALPHA;
        i += stride; // Move to next batch
    }
}