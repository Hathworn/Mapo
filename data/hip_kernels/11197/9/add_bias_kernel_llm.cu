#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int n, int size)
{
    // Calculate the global index for each thread
    int idx = blockIdx.z * gridDim.y * size + blockIdx.y * size + blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check bounds to ensure we are within size limits
    if(idx < n * size) {
        int filter = (idx / size) % n;
        output[idx] += biases[filter];  // Add bias to the appropriate position
    }
}