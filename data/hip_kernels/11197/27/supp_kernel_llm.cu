#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified grid calculation
    if(i < N) {
        float val = X[i * INCX]; // Cache value to reduce repetitive accesses
        if(val * val < ALPHA * ALPHA) {
            X[i * INCX] = 0;
        }
    }
}