#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Ensure thread is within bounds
    if (i < n) {
        // Use local variable to avoid multiple memory reads
        float dc_val = dc[i];
        float s_val = s[i];

        if (da) {
            // Use fused multiply-add operation for possible performance benefit
            da[i] = fmaf(dc_val, s_val, da[i]);
        }
        
        db[i] = fmaf(dc_val, 1.0f - s_val, db[i]);
        ds[i] = fmaf(dc_val, a[i], ds[i]) - dc_val * b[i];  // Reuse dc_val for common subexpression
    }
}