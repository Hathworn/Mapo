#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
#define BLOCK 1024

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0;

    // Loop over batch and size in parallel
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Using a single thread to accumulate results
    if (p == 0) {
        float total_sum = 0;
        for (int i = 0; i < BLOCK; ++i) {
            total_sum += part[i];
        }
        bias_updates[filter] += total_sum;
    }
}