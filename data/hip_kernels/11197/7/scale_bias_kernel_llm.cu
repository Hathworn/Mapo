#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    int filter = blockIdx.y;
    int batch = blockIdx.z;
    int base_index = (batch * n + filter) * size;

    // Use block-level parallelism for increased efficiency
    for (int offset = blockIdx.x * blockDim.x + threadIdx.x; offset < size; offset += blockDim.x * gridDim.x) {
        output[base_index + offset] *= biases[filter];
    }
}