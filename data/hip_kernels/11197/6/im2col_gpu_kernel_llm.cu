#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void im2col_gpu_kernel(const int n, const float* data_im, const int height, const int width, const int ksize, const int pad, const int stride, const int height_col, const int width_col, float *data_col) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a while loop for better control over the iteration and to avoid redundant operations
    while (index < n) {
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        
        float* data_col_ptr = data_col + (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im + (channel_in * height + h_in) * width + w_in;
        
        // Use a single loop with index checks to reduce control overhead
        #pragma unroll
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;
                
                // Conditional assignment for clarity and reduced branching
                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ? data_im_ptr[i * width + j] : 0;
                
                // Move the pointer directly inside the loop to reduce calculations outside
                data_col_ptr += height_col * width_col;
            }
        }
        
        // Increment index within the while loop for continuous checking
        index += blockDim.x * gridDim.x;
    }
}