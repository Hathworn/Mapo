#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate the thread's global index using blockIdx, blockDim, and threadIdx
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    // Check to ensure index is within bounds
    if (i < n) {
        // Compute the weighted sum with branchless logic for b
        float b_val = (b != nullptr) ? b[i] : 0.0f;
        c[i] = s[i] * a[i] + (1.0f - s[i]) * b_val;
    }
}