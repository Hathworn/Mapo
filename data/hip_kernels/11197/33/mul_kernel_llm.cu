#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Optimize index calculation for 1D grid
    int gridSize = blockDim.x * gridDim.x; // Calculate grid size
    while (i < N) {
        Y[i * INCY] *= X[i * INCX];
        i += gridSize; // Stride through the array
    }
}