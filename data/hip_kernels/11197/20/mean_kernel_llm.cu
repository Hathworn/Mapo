#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized mean_kernel function
__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean) {
    float scale = 1.0f / (batch * spatial);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float sum = 0.0f; // Use a local variable for accumulation
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            int index = j * filters * spatial + i * spatial + k;
            sum += x[index]; // Use the local variable for accumulation
        }
    }
    mean[i] = sum * scale;
}