#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;

    // Use shared memory to improve performance
    __shared__ float mean_shared;
    float local_mean = 0.0f;

    // Reduce memory access by eliminating redundant calculations
    for (int i = 0; i < n; ++i) {
        local_mean += fabsf(input[i * size + s]);
    }
    
    // Calculate the mean once for the entire thread block
    if (threadIdx.x == 0) {
        mean_shared = local_mean / n;
    }
    __syncthreads();

    // Use the pre-computed mean for binarization
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean_shared : -mean_shared;
    }
}