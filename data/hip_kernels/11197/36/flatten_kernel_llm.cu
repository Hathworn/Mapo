#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    // Calculate the global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Return early if out of bounds
    if (i >= N) return;

    // Calculate indices using modulo and division
    int in_s = i % spatial;
    int in_c = (i / spatial) % layers;
    int b = i / (spatial * layers);

    // Calculate the input and output indices
    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;

    // Condition to determine direction of operation
    out[forward ? i2 : i1] = x[forward ? i1 : i2];
}