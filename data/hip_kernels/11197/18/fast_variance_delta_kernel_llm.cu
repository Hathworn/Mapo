#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    float local_sum = 0.0f; // Use a register to accumulate local results

    for(int j = 0; j < batch; ++j){
        for(int i = id; i < spatial; i += threads){ // Increment by threads to ensure each thread processes different elements
            int index = j * spatial * filters + filter * spatial + i;
            local_sum += delta[index] * (x[index] - mean[filter]);
        }
    }

    local[id] = local_sum; // Store the result in shared memory
    __syncthreads();

    if(id == 0){
        float sum = 0.0f;
        for(int i = 0; i < threads; ++i){
            sum += local[i];
        }
        variance_delta[filter] = sum * -.5f * powf(variance[filter] + .00001f, -1.5f);
    }
}