#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use shared memory for fast access in threads within a block
    extern __shared__ float s_pred[];
    extern __shared__ float s_truth[];
    
    if (i < n) {
        s_pred[threadIdx.x] = pred[i];
        s_truth[threadIdx.x] = truth[i];
        __syncthreads();  // Synchronize threads within a block

        float diff = s_truth[threadIdx.x] - s_pred[threadIdx.x];
        error[i] = fabsf(diff);  // Use fabsf for performance
        delta[i] = copysignf(1.0f, diff);  // Use copysign for simplicity
    }
}