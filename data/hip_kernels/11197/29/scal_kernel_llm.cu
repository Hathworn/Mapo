#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    /* Calculate global thread index */
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    /* Unroll the loop for better performance */
    int stride = blockDim.x * gridDim.x * blockDim.y;
    for (; i < N; i += stride)
    {
        X[i * INCX] *= ALPHA;
    }
}