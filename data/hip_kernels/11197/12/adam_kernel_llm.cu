#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t) {
    // Optimize index calculation using 3D thread and block values
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) {
        // Cache repeated calculations to improve performance
        float B1_t = pow(B1, t);
        float B2_t = pow(B2, t);
        float rate_adjusted = rate * sqrt(1.0f - B2_t) / (1.0f - B1_t);
        
        x[index] += rate_adjusted * m[index] / (sqrt(v[index]) + eps);
    }
}