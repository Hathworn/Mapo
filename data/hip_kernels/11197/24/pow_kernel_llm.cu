#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Calculate thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    
    // Ensure within bounds
    if (i < N) {
        // Calculate index based on increments and apply operation
        int indexX = i * INCX; 
        int indexY = i * INCY; 
        Y[indexY] = powf(X[indexX], ALPHA); // Use powf for float precision
    }
}