#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = blockIdx.z * (gridDim.y * gridDim.x * blockDim.x) + blockIdx.y * (gridDim.x * blockDim.x) + blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -(size / 2);
    int h_offset = -(size / 2);

    int out_index = j + w * (i + h * (k + c * b));
    float local_delta = 0.0f; // Accumulate delta locally to reduce memory access
    for(int l = 0; l < size; ++l) {
        for(int m = 0; m < size; ++m) {
            int cur_h = h_offset + i + l;
            int cur_w = w_offset + j + m;
            int index = cur_w + w * (cur_h + h * (k + b * c));
            if (cur_h >= 0 && cur_h < h && cur_w >= 0 && cur_w < w) {
                 local_delta += rate * (x[index] - x[out_index]); // Use local_delta
            }
        }
    }
    atomicAdd(&delta[out_index], local_delta); // Atomic update to prevent race condition
}