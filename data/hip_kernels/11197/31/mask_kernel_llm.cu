#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask)
{
    // Calculate the global thread ID more efficiently
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use a loop to handle scenarios where grid size isn't a multiple of n
    for (; i < n; i += blockDim.x * gridDim.x) {
        // Simplified condition check and assignment
        if (mask[i] == mask_num) {
            x[i] = mask_num;
        }
    }
}