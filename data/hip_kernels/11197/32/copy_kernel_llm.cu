#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Use 1D grid for simplicity
    int gridSize = gridDim.x * blockDim.x; // Calculate total grid size

    for (; i < N; i += gridSize) { // Loop over elements in increments of gridSize
        Y[i * INCY + OFFY] = X[i * INCX + OFFX]; // Efficient memory access
    }
}