#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + 2*pad)/stride;
    int w = (in_w + 2*pad)/stride;
    int c = in_c;
    int area = (size-1)/stride;

    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    const int index = id;
    int j = index % in_w;
    int i = (index / in_w) % in_h;
    int k = (index / (in_w * in_h)) % in_c;
    int b = index / (in_c * in_h * in_w);

    int w_offset = -pad;
    int h_offset = -pad;

    float d = 0.0f; // initialization using float f
    #pragma unroll // Unroll the loops for better parallelism
    for(int l = -area; l <= area; ++l) {
        for(int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset)/stride + m;
            int out_h = (i - h_offset)/stride + l;
            int out_index = out_w + w * (out_h + h * (k + c * b));
            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h) { // Combine condition check
                d += (indexes[out_index] == index) ? delta[out_index] : 0.0f;
            }
        }
    }
    prev_delta[index] += d;
}