#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;
    
    int k = id % c;
    int b = id / c;
    
    float sum = 0.0f; // Accumulate sum as a local variable
    
    for (int i = 0; i < w * h; ++i) {
        int in_index = i + h * w * (k + b * c);
        sum += input[in_index]; // Accumulate input values
    }
    
    output[k + c * b] = sum / (w * h); // Compute average directly to output
}