#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate the linear thread index to efficiently access elements
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = gridDim.x * blockDim.x;
    
    // Efficiently loop through elements in steps of grid stride
    for (; i < N; i += gridStride) 
    {
        X[i * INCX] = ALPHA;
    }
}