#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void block_sum(const float *input, float *per_block_results, const size_t n)
{
    extern __shared__ float sdata[];

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input into __shared__ memory
    if(i < n)
    {
        sdata[threadIdx.x] = input[i];
    }
    else
    {
        sdata[threadIdx.x] = 0.0f; // Initialize with 0 when out of bounds
    }
    __syncthreads();

    // Unrolled loop for contiguous range pattern
    for(int offset = blockDim.x / 2; offset > 32; offset >>= 1)
    {
        if(threadIdx.x < offset)
        {
            sdata[threadIdx.x] += sdata[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Optimize the last warp without barriers
    if(threadIdx.x < 32)
    {
        volatile float *vsdata = sdata;
        vsdata[threadIdx.x] += vsdata[threadIdx.x + 32];
        vsdata[threadIdx.x] += vsdata[threadIdx.x + 16];
        vsdata[threadIdx.x] += vsdata[threadIdx.x + 8];
        vsdata[threadIdx.x] += vsdata[threadIdx.x + 4];
        vsdata[threadIdx.x] += vsdata[threadIdx.x + 2];
        vsdata[threadIdx.x] += vsdata[threadIdx.x + 1];
    }

    // Thread 0 writes the final result
    if(threadIdx.x == 0)
    {
        per_block_results[blockIdx.x] = sdata[0];
    }
}