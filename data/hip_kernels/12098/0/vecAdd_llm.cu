#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(double *a, double *b, double *c, int n) 
{
    // Calculate global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    // Use warp-based optimization by processing multiple elements if possible
    int stride = blockDim.x * gridDim.x;
    for (int i = id; i < n; i += stride) {
        c[i] = a[i] * b[i];
    }
}