#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use a thread-safe atomic operation to prevent race conditions
    atomicAdd(accuracy, *accuracy / N);
}