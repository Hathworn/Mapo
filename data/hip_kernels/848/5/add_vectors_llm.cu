#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_vectors(float *ad, float *bd, int N)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    // Use if-condition only when necessary
    if (index < N) 
    {
        ad[index] += bd[index]; // Optimization: Direct addition within the conditional block
    }
}