#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_array(float *ad, int N)
{
    // Utilize shared memory for faster access
    extern __shared__ float shared_ad[];
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int localIdx = threadIdx.x;

    // Load data into shared memory
    if (index < N) {
        shared_ad[localIdx] = ad[index];
        __syncthreads();  // Synchronize to ensure all loads are complete

        // Square the values in shared memory
        shared_ad[localIdx] *= shared_ad[localIdx];

        // Write back to global memory
        ad[index] = shared_ad[localIdx];
    }
}