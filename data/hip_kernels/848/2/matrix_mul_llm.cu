#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void matrix_mul(float *ad, float *bd, float *cd, int N) {
    // Use shared memory for faster access
    __shared__ float ad_shared[TILE_SIZE][TILE_SIZE];
    __shared__ float bd_shared[TILE_SIZE][TILE_SIZE];

    // Compute row and column for each thread
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    float pvalue = 0;

    // Loop through tiles of the matrices
    for (int t = 0; t < N / TILE_SIZE; ++t) {
        // Load data into shared memory
        ad_shared[threadIdx.y][threadIdx.x] = ad[Row * N + t * TILE_SIZE + threadIdx.x];
        bd_shared[threadIdx.y][threadIdx.x] = bd[(t * TILE_SIZE + threadIdx.y) * N + Col];
        
        // Synchronize threads in block after loading data
        __syncthreads();

        // Compute partial product for the tile
        for (int i = 0; i < TILE_SIZE; ++i) {
            pvalue += ad_shared[threadIdx.y][i] * bd_shared[i][threadIdx.x];
        }

        // Synchronize threads to ensure calculation is done before loading new data
        __syncthreads();
    }

    // Write result to the output matrix
    cd[Row * N + Col] = pvalue;
}
```
