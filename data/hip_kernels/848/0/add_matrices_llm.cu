#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_matrices(float *ad, float *bd, float *cd, int N)
{
    // Calculate row and column indices
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure that the indices are within bounds
    if (row < N && col < N)
    {
        // Perform element-wise addition
        cd[row * N + col] = ad[row * N + col] + bd[row * N + col];
    }
}