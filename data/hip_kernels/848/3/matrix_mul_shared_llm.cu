#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Macro for tile size, ensures flexibility
#define TILE 16

__global__ void matrix_mul_shared(float *ad, float *bd, float *cd, int N)
{
    float pvalue = 0;

    int ty = threadIdx.y;
    int tx = threadIdx.x;

    // Allocate shared memory per block
    __shared__ float ads[TILE][TILE];
    __shared__ float bds[TILE][TILE];

    // Find Row and Column corresponding to a data element for each thread
    int Row = blockIdx.y * TILE + ty;
    int Col = blockIdx.x * TILE + tx;

    // Iterate through TILEs to traverse the whole width
    for (int i = 0; i < N / TILE; ++i)
    {
        // Copy values of data TILE into shared memory
        ads[ty][tx] = ad[Row * N + (i * TILE) + tx];
        bds[ty][tx] = bd[(i * TILE + ty) * N + Col];

        __syncthreads(); // Synchronize to confirm that the whole TILE has been copied

        // Calculate partial dot-product
        #pragma unroll
        for (int k = 0; k < TILE; k++)
            pvalue += ads[ty][k] * bds[k][tx];

        __syncthreads(); // Synchronize to confirm that the whole partial product has been calculated
    }

    // Store dot product at the corresponding position in the resultant Matrix
    cd[Row * N + Col] = pvalue;
}