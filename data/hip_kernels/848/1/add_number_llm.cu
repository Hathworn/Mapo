#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_number(float *ad, float *bd)
{
    // Use shared memory for fast access
    __shared__ float shared_ad;
    __shared__ float shared_bd;

    if (threadIdx.x == 0) {
        shared_ad = *ad;
        shared_bd = *bd;
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        shared_ad += shared_bd;
        *ad = shared_ad; // Update global memory after addition
    }
}