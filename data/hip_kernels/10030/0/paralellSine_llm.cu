#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// problem size (vector length) N
static const int N = 12345678;

// Number of terms to use when approximating sine
static const int TERMS = 6;

// kernel function (CPU - Do not modify)
__global__ void paralellSine(float *input, float *output)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x; // Proper indexing of elements
    if (idx >= N) return; // Boundary check to prevent invalid memory access

    float value = input[idx];
    float numer = value * value * value;
    int denom = 6;
    int sign = -1;

    #pragma unroll // Unroll loop for better performance
    for (int j = 1; j <= TERMS; j++)
    {
        value += sign * numer / denom;
        numer *= value * value;
        denom *= (2 * j + 2) * (2 * j + 3);
        sign *= -1;
    }
    output[idx] = value;
}