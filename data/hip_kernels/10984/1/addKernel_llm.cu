#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    c[index] = a[index] + b[index];
}