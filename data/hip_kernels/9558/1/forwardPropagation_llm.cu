#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forwardPropagation(float *a_d, float *b_d, int size)
{
    int idx = threadIdx.x;
    int idy = threadIdx.y;
    int global_idx = idy * (size + 1) + idx;

    __shared__ float temp[16][16];

    // Load data into shared memory
    temp[idy][idx] = a_d[global_idx];
    __syncthreads(); // Ensure shared memory is updated before proceeding

    // Optimize loop by reducing shared memory accesses and unnecessary computations
    for (int i = 1; i < size; i++)
    {
        if ((idy + i) < size)
        {
            float var1 = -temp[i-1][i-1] / temp[i+idy][i-1];
            // Use single shared memory access per thread iteration
            temp[i+idy][idx] += var1 * temp[i+idy][idx];
        }
        __syncthreads(); // Ensure all threads have finished before next iteration
    }

    // Store result back to global memory
    b_d[global_idx] = temp[idy][idx];
}