#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_div(char* newB, char* first, char* second, int size_first, int size_second, int * size_newB, char* aux) {
    int i = threadIdx.x;
    int j = threadIdx.y;
    
    // Handle sign in the first thread
    if (j == 0 && i == 0) {
        if (first[j] == '-' || second[i] == '-') {
            newB[0] = '-';
        } else {
            newB[0] = '+';
        }
        return;
    }

    // Perform initial subtraction, using shared memory for efficiency
    extern __shared__ int shared_aux[];
    int diff = size_first - size_second;
    int tmp = 0;
    
    if (j - 1 - diff >= 0 && (second[j - 1 - diff] != '+' && second[j - 1 - diff] != '-')) {
        tmp = first[j - 1] - second[j - 1 - diff];
    } else if (first[j - 1] != '+' && first[j - 1] != '-') {
        tmp = first[j - 1];
    }

    if (tmp < 0) {
        aux[i * size_first + j - 1]--;
        tmp += 10;
    }

    if (i != 0) {
        shared_aux[i * size_first + j] = aux[i * size_first + j] + tmp;
    } else {
        shared_aux[j] = tmp;
    }
    
    __syncthreads();  // Ensure all shared memory updates are complete

    aux[i * size_first + j] = shared_aux[i * size_first + j];  // Update global memory from shared memory
}