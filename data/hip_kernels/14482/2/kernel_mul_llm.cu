#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_mul(char* newB, char* first, char* second, int size_first, int size_second, int* size_newB) {
    // Use blockIdx.x and blockIdx.y for grid-based indexing
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    int tid = j * gridDim.x * blockDim.x + i;

    // Bounds check to ensure valid accesses
    if (i < size_first && j < size_second) {
        if (j != 0 || i != 0) {  // Corrected logic for first element
            newB[tid] = first[i] * second[j];
        } else {
            newB[0] = (first[j] != second[i]) ? '-' : '+';
        }
    }
}