#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline void charAtomicAdd(char *address, char value) {
    int oldval, newval, readback;
    oldval = *address;
    newval = oldval + value;
    while ((readback = atomicCAS((int *)address, oldval, newval)) != oldval) {
        oldval = readback;
        newval = oldval + value;
    }
}

__global__ void kernel_add(char* newB, char* first, char* second, int size_biggest, int diff, int* size_newB) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;  // Use global index for larger grid sizes
    if (i >= size_biggest) return;  // Avoid out-of-bounds access

    int tmp = 0;
    
    if (i == 0 || i - 1 - diff < 0 || (second[i - 1 - diff] == '+' || second[i - 1 - diff] == '-')) {
        if (first[i - 1] != '+' && first[i - 1] != '-') {
            tmp = first[i - 1];
        }
    } else {
        tmp = second[i - 1 - diff] + first[i - 1];
    }

    if (tmp >= 10) {
        atomicAdd(&newB[i - 1], 1);  // Use atomicAdd for correctness
        tmp %= 10;
    }

    atomicAdd(&newB[i], tmp);  // Use atomicAdd for concurrent writes correction
}