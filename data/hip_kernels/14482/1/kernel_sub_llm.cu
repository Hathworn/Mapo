#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void kernel_sub(char* newB, char* first, char* second, int size_biggest, int diff, int * size_newB) {
    int i = threadIdx.x;  // Simplified thread index retrieval
    if (i == 0) return;   // Skip processing for thread 0

    int tmp = 0;
    // Eliminate redundant loop: Threads handle one element each

    // Optimize boundary check and computation
    if (i - 1 - diff >= 0) {
        if (second[i - 1 - diff] != '+' && second[i - 1 - diff] != '-') {
            tmp = first[i - 1] - second[i - 1 - diff];
        }
    } else if (first[i - 1] != '+' && first[i - 1] != '-') {
        tmp = first[i - 1];
    }

    if (tmp < 0) {
        newB[i - 1]--;  // Simplified to directly decrement
        tmp += 10;      // Adjustment for negative tmp
    }
    
    newB[i] += tmp;     // Combine non-zero check into main operation
}