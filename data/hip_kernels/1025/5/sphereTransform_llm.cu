#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sphereTransform(float *data, const unsigned int N)
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure threads are within bounds and coalesce data access
    if (idx < N)
    {
        float temp = data[idx];
        data[idx] = temp * 360.0f - 180.0f;      // Transformation for first half of data
        data[idx + N] = acosf(2.0f * data[idx + N] - 1.0f); // Transformation for second half
    }
}