#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sinewave(float *heightMap, unsigned int width, unsigned int height, float time)
{
    const float freq = 4.0f;

    // Pre-calculate values used multiple times
    const float invWidth = 2.0f / width;
    const float invHeight = 2.0f / height;

    // Iterate through the entire array in a way that is independent of the grid configuration
    for (size_t tid = blockIdx.x * blockDim.x + threadIdx.x; tid < width * height; tid += blockDim.x * gridDim.x) {
        // Calculate the x, y coordinates
        const size_t y = tid / width;
        const size_t x = tid % width;

        // Normalize x, y to [-1,1]
        const float u = x * invWidth - 1.0f;
        const float v = y * invHeight - 1.0f;

        // Calculate the new height value and store it
        heightMap[tid] = 0.5f * sinf(u * freq + time) * cosf(v * freq + time);
    }
}