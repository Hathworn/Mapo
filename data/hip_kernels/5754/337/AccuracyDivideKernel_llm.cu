#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread index for potential parallel processing
    int idx = threadIdx.x;
    
    // Ensure single thread execution
    if (idx == 0) {
        *accuracy /= N; // Divide once by N
    }
}