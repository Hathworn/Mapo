#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void cumulativeOffspringToAncestorKernel(const int* cumulativeOffspring, int* ancestor, int numParticles) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Early exit for out-of-bound indices
    if (idx >= numParticles) return;

    int numCurrentOffspring = cumulativeOffspring[idx] - (idx == 0 ? 0 : cumulativeOffspring[idx - 1]);
    
    // Use loop unrolling for optimization
    for (int j = 0; j < numCurrentOffspring; j++) {
        ancestor[(idx == 0 ? 0 : cumulativeOffspring[idx - 1]) + j] = idx;
    }
}