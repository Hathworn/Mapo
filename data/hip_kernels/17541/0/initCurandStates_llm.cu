#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

/*
* File smc_kernels.cu contains definitions of kernels used by SMC.
*/

#ifdef __NVCC__




#endif

__global__ void initCurandStates(hiprandState* randStates, int numThreads, int seed) {
    // Use grid-stride loop for better scalability and efficiency
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numThreads; i += blockDim.x * gridDim.x) {
        // Directly initialize local state variable
        hiprandState randStateLocal;
        hiprand_init(1234 + clock64(), seed * numThreads + i, 0, &randStateLocal);
        randStates[i] = randStateLocal;
    }
}