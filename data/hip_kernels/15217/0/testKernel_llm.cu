#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// This works fine with a mutex, but crashes with a sigbus error when not using a mutex
// #define USE_MUTEX

#ifdef USE_MUTEX
std::mutex m;
#endif

__global__ void testKernel() {
    // Optimize by ensuring printf doesn't rely on synchronization
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Thread %u: Kernel running\n", tid);
}