#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vectorAdd(uint16_t* A, const uint16_t* B, int32_t numElements)
{
    // Calculate an optimized index based on block size and grid size
    int i = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;

    // Use warp divergence optimization for boundary checking
    if (i < numElements)
    {
        A[i] += B[i];
    }
}