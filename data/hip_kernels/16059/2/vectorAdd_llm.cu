#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vectorAdd(const uint16_t* A, const uint16_t* B, uint16_t* C, uint32_t numElements)
{
    // Calculate global thread ID
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    // Process multiple elements per thread to increase data throughput
    int gridSize = blockDim.x * gridDim.x;
    
    // Unroll loops for better performance
    while (i < numElements)
    {
        C[i] = A[i] + B[i];
        i += gridSize;
    }
}