#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Using empty __global__ function to avoid compilation errors
// optimized with no redundant code or logic
__global__ void KernelVersionShim() { }