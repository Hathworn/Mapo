#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void blend_kernel(float *outSrc, const float *inSrc) {
    // map from threadIdx/BlockIdx to pixel position
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // Use shared memory for coalesced memory access
    extern __shared__ float s_inSrc[];

    // Load data to shared memory: Each thread loads one element
    int local_x = threadIdx.x + 1; // Avoid boundary condition
    int local_y = threadIdx.y + 1; // Avoid boundary condition
    int local_offset = local_x + local_y * (blockDim.x + 2);

    // Load central, left, right, top, and bottom pixels into shared memory
    s_inSrc[local_offset] = inSrc[offset];

    // Load boundary pixels
    if (threadIdx.x == 0) s_inSrc[local_offset - 1] = inSrc[offset - 1];
    if (threadIdx.x == blockDim.x - 1) s_inSrc[local_offset + 1] = inSrc[offset + 1];
    if (threadIdx.y == 0) s_inSrc[local_offset - (blockDim.x + 2)] = inSrc[offset - DIM];
    if (threadIdx.y == blockDim.y - 1) s_inSrc[local_offset + (blockDim.x + 2)] = inSrc[offset + DIM];

    __syncthreads();

    // Compute new value
    float center = s_inSrc[local_offset];
    float result = center + SPEED * (s_inSrc[local_offset - 1] +
                                     s_inSrc[local_offset + 1] +
                                     s_inSrc[local_offset - (blockDim.x + 2)] +
                                     s_inSrc[local_offset + (blockDim.x + 2)] -
                                     center * 4);

    // Update output array
    outSrc[offset] = result;
}