#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void copy_const_kernel(float *iptr, const float *cptr) {
    // Calculate the global thread index
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int offset = x + y * blockDim.x * gridDim.x;

    // Perform copy only if condition is met
    if (cptr[offset] != 0.0f) {
        iptr[offset] = cptr[offset];
    }
}