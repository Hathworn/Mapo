#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sumArraysOnGPU(float *A, float *B, float *C)
{
    // Optimization: Use shared memory to store block data
    extern __shared__ float sharedA[], sharedB[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    // Load data into shared memory
    sharedA[tid] = A[i];
    sharedB[tid] = B[i];
    __syncthreads();

    // Perform computation using shared memory
    C[i] = sharedA[tid] + sharedB[tid];
}