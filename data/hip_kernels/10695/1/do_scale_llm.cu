#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void do_scale(uint8_t * inBuffer, uint8_t * outBuffer, uint32_t inWidth, uint32_t inHeight, uint32_t outWidth, uint32_t outHeight, double ratioHeight, double ratioWidth, double xRatio, double yRatio, uint32_t outHeightOffset, uint32_t outWidthOffset, uint32_t inStep, uint32_t outStep)
{
    // Calculate output indices based on block and thread indices
    uint32_t outRowIndex = blockIdx.x + outHeightOffset;
    uint32_t outColIndex = threadIdx.x + outWidthOffset;
    uint32_t outIndex = (outRowIndex) * outWidth + outColIndex;

    // Calculate input indices for interpolation
    uint32_t inX00row = min(static_cast<uint32_t>(outRowIndex / ratioHeight), inHeight - 2); // Ensure within bounds
    uint32_t inX00col = min(static_cast<uint32_t>(outColIndex / ratioWidth), inWidth - 2);  // Ensure within bounds

    // Load input buffer values for bilinear interpolation
    uint8_t inX00 = inBuffer[(inX00row * inWidth + inX00col) * inStep];
    uint8_t inX01 = inBuffer[(inX00row * inWidth + inX00col + 1) * inStep];
    uint8_t inX10 = inBuffer[((inX00row + 1) * inWidth + inX00col) * inStep];
    uint8_t inX11 = inBuffer[((inX00row + 1) * inWidth + inX00col + 1) * inStep];

    // Perform bilinear interpolation and store result in output buffer
    outBuffer[outIndex * outStep] = static_cast<uint8_t>(
        yRatio * (xRatio * inX00 + (1 - xRatio) * inX01) +
        (1 - yRatio) * (xRatio * inX10 + (1 - xRatio) * inX11)
    );
}