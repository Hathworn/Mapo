#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ static void update_inverse_cuda(float *Ainv, float *u, int N, int rowstride, int k) {
    __shared__ float A_k[NMAX], u_shared[NMAX], Ainv_u[NMAX], Ainv_shared[NMAX];

    // Load A_k and u_shared into shared memory
    A_k[threadIdx.x] = Ainv[k * rowstride + threadIdx.x];
    u_shared[threadIdx.x] = u[threadIdx.x];

    // Compute k'th element of Ainv_u using block-wide reduction
    Ainv_u[threadIdx.x] = u_shared[threadIdx.x] * A_k[threadIdx.x];
    __syncthreads();
    for (int n = N >> 1; n > 0; n >>= 1) {
        if (threadIdx.x < n) {
            Ainv_u[threadIdx.x] += Ainv_u[threadIdx.x + n];
        }
        __syncthreads();
    }
    
    float prefact = -1.0f / (1.0f + Ainv_u[0]);

    for (int row = 0; row < N; row++) {
        // Load Ainv_shared into shared memory
        Ainv_shared[threadIdx.x] = Ainv[row * rowstride + threadIdx.x];
        __syncthreads();

        // Compute row'th element of Ainv_u using block-wide reduction
        Ainv_u[threadIdx.x] = u_shared[threadIdx.x] * Ainv_shared[threadIdx.x];
        for (int n = N >> 1; n > 0; n >>= 1) {
            if (threadIdx.x < n) {
                Ainv_u[threadIdx.x] += Ainv_u[threadIdx.x + n];
            }
            __syncthreads();
        }
        __syncthreads();

        // Update Ainv with computed result
        Ainv[row * rowstride + threadIdx.x] =
            Ainv_shared[threadIdx.x] + prefact * Ainv_u[0] * A_k[threadIdx.x];
    }
}