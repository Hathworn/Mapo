#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel(float *a, size_t N)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    __shared__ float s[BS];
    float sum = 0.0f;
    
    int off = bid * blockDim.x + tid;
    if (off < N) {
        s[tid] = a[off];
    } else {
        s[tid] = 0.0f; // Avoid out-of-bound access
    }
    __syncthreads();
    
    // Optimize reduction to avoid bank conflicts
    for (int skip = blockDim.x / 2; skip > 0; skip >>= 1) {
        if (tid < skip && tid + skip < N) {
            s[tid] += s[tid + skip];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        atomicAdd(&a[0], s[0]); // Use atomic operation to accumulate result
    }
}