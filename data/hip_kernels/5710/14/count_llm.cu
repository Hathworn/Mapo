#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void count(int *data, int input, int *result)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < gridDim.x * blockDim.x) // Ensure within bounds
    {
        if (data[i] == input)
        {
            atomicAdd(result, 1); // Direct atomicAdd without a temporary variable
        }
    }
}