#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecMultiplyReverse(int *A, int *B, int *C)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use ternary operator for concise condition handling
    C[i] = (i % 2 == 0) ? (A[i] + B[i]) : (A[i] - B[i]);
}