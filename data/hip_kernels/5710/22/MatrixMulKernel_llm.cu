#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulKernel (float* Md, float* Nd, float* Pd, int ncols) {
    // Shared memory for Md and Nd sub-matrices
    __shared__ float Ms[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Ns[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float Pvalue = 0;

    // Loop over sub-matrices for tiled multiplication
    for (int m = 0; m < (ncols / BLOCK_SIZE); ++m) {
        // Load Md and Nd sub-matrices into shared memory
        Ms[threadIdx.y][threadIdx.x] = Md[row * ncols + (m * BLOCK_SIZE + threadIdx.x)];
        Ns[threadIdx.y][threadIdx.x] = Nd[(m * BLOCK_SIZE + threadIdx.y) * ncols + col];
        __syncthreads();

        // Multiply Md and Nd sub-matrices
        for (int k = 0; k < BLOCK_SIZE; ++k) {
            Pvalue += Ms[threadIdx.y][k] * Ns[k][threadIdx.x];
        }
        __syncthreads();
    }

    // Write back the result
    Pd[row * ncols + col] = Pvalue;
}