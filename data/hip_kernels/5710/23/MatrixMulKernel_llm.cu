#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulKernel(float* Md, float* Nd, float* Pd, int ncols) {
    // Calculate the row and column indices for the current thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize the output element to zero
    float Pvalue = 0;

    // Ensure thread is within matrix bounds
    if (row < ncols && col < ncols) {
        // Compute the matrix multiplication result for the thread's position
        for (int k = 0; k < ncols; k++) {
            Pvalue += Md[row * ncols + k] * Nd[k * ncols + col];
        }
        // Store the result in the output matrix
        Pd[row * ncols + col] = Pvalue;
    }
}