#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(int *A, int *B, int *C) {
    // Calculate index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure within bounds
    if (i < N) {
        // Perform addition
        C[i] = A[i] + B[i];
    }
}