#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16 // Assuming TILE_WIDTH is defined here

__global__ void matrixMul(float* A, float* B, float* C, int width)
{
    __shared__ float As[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];

    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    float c_val = 0.0f;

    // Loop over each sub-matrix required to compute Csub
    for(int i = 0; i < width / TILE_WIDTH; ++i)
    {
        // Load tiles into shared memory
        int a_index = row * width + i * TILE_WIDTH + threadIdx.x;
        int b_index = (i * TILE_WIDTH + threadIdx.y) * width + col;

        As[threadIdx.y][threadIdx.x] = A[a_index];
        Bs[threadIdx.y][threadIdx.x] = B[b_index];

        __syncthreads();

        // Compute dot product for current tile
        #pragma unroll
        for(int k = 0; k < TILE_WIDTH; ++k)
        {
            c_val += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    // Write the result
    int c_index = row * width + col;
    C[c_index] = c_val;
}