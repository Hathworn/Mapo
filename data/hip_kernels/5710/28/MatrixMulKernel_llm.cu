#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulKernel (float* Md, float* Nd, float* Pd, int ncols) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if the thread is within matrix boundaries
    if (row < ncols && col < ncols) {
        float Pvalue = 0.0f;
        // Use shared memory for faster access
        __shared__ float Mds[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float Nds[BLOCK_SIZE][BLOCK_SIZE];

        for (int k = 0; k < (ncols + BLOCK_SIZE - 1) / BLOCK_SIZE; ++k) {
            // Load data into shared memory
            if (row < ncols && (k * BLOCK_SIZE + threadIdx.x) < ncols) {
                Mds[threadIdx.y][threadIdx.x] = Md[row * ncols + k * BLOCK_SIZE + threadIdx.x];
            } else {
                Mds[threadIdx.y][threadIdx.x] = 0.0f;
            }
            
            if (col < ncols && (k * BLOCK_SIZE + threadIdx.y) < ncols) {
                Nds[threadIdx.y][threadIdx.x] = Nd[(k * BLOCK_SIZE + threadIdx.y) * ncols + col];
            } else {
                Nds[threadIdx.y][threadIdx.x] = 0.0f;
            }
            
            // Ensure all threads have loaded their segment into shared memory
            __syncthreads();

            for (int n = 0; n < BLOCK_SIZE; ++n) {
                Pvalue += Mds[threadIdx.y][n] * Nds[n][threadIdx.x];
            }
            
            // Ensure all threads are done computing using current shared memory segment
            __syncthreads();
        }
        
        Pd[row * ncols + col] = Pvalue;
    }
}