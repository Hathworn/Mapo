#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulKernel (float* Md, float* Nd, float* Pd, int ncols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < ncols && col < ncols) { // Ensure threads do not access out of bounds memory
        float Pvalue = 0.0f;
        for (int k = 0; k < ncols; ++k) {
            float Melement = Md[row * ncols + k];
            float Nelement = Nd[k * ncols + col];
            Pvalue += Melement * Nelement;
        }
        Pd[row * ncols + col] = Pvalue;
    }
}