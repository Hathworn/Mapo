#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void reverseArray(int *A, int *B) {
    int threadID = threadIdx.x;
    // Calculate the number of elements this thread will handle
    int elementsPerThread = ArraySize / blockDim.x;
    int start = threadID * elementsPerThread;
    int end = start + elementsPerThread - 1;

    // Use a more efficient loop structure with single comparison
    for (int i = 0; i < elementsPerThread; i++) {
        B[end - i] = A[start + i];
    }
}