#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function with boundary check
__global__ void vecAdd(float *A, float *B, float *C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];  // Ensure thread does not exceed array bounds
    }
}