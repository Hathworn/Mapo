#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define TILE_WIDTH 16 // Define TILE_WIDTH if not already defined

__global__ void matrixMul(float* A, float* B, float* C, int width)
{
    __shared__ float As[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Bs[TILE_WIDTH][TILE_WIDTH];
    
    int row = blockIdx.y * TILE_WIDTH + threadIdx.y;
    int col = blockIdx.x * TILE_WIDTH + threadIdx.x;
    float c_val = 0.0f;
    
    for(int i = 0; i < width/TILE_WIDTH; i++)
    {
        // Load matrix elements from global memory into shared memory
        As[threadIdx.y][threadIdx.x] = A[row * width + (i * TILE_WIDTH + threadIdx.x)];
        Bs[threadIdx.y][threadIdx.x] = B[(i * TILE_WIDTH + threadIdx.y) * width + col];
        __syncthreads(); // Synchronize to ensure all threads have loaded their data

        // Perform matrix multiplication within a tile
        #pragma unroll
        for(int k = 0; k < TILE_WIDTH; k++)
        {
            c_val += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }
        
        __syncthreads(); // Synchronize before loading new tile
    }
    
    // Store the result into global memory
    C[row * width + col] = c_val;
}