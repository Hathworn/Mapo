#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecAdd(int *A, int *B, int *C, int numElements) {
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure thread index does not exceed number of elements
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}