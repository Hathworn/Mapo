#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void grayscale(float4* imagem, int width, int height)
{
    // Calculate the unique thread index using 2D grid and block indices
    const int i = blockIdx.y * blockDim.y * width + blockIdx.x * blockDim.x
                + threadIdx.y * width + threadIdx.x;

    if (i < width * height)
    {
        // Compute grayscale value
        float v = 0.3f * imagem[i].x + 0.6f * imagem[i].y + 0.1f * imagem[i].z;
        // Assign grayscale value to all color channels
        imagem[i] = make_float4(v, v, v, imagem[i].w);
    }
}