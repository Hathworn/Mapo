#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Ensure only one thread performs the division
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}