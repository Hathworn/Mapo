#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void uniformAdd(int *g_data, int *uniforms, int n, int blockOffset, int baseIndex) {
    __shared__ float uni;
    // Load shared memory once per block
    if (threadIdx.x == 0) {
        uni = uniforms[blockIdx.x + blockOffset];
    }

    unsigned int address = __mul24(blockIdx.x, (blockDim.x << 1)) + baseIndex + threadIdx.x;

    __syncthreads();

    // Add shared value to two positions in global memory
    int offset = threadIdx.x + blockDim.x;
    if (address < n) {
        g_data[address] += uni;
    }
    if (offset < n) {
        g_data[address + blockDim.x] += uni;
    }
}