#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void ComputeOffsetOfMatrixA(const int32_t* col_sum, int32_t* output, int32_t N) {
    // Calculate the index for each thread
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use a stride loop to ensure access across the entire array
    for (int32_t i = idx; i < blockIdx.x * N + N; i += blockDim.x * gridDim.x) {
        output[i] = -col_sum[i % N]; // Use modulo to correctly index col_sum
    }
}