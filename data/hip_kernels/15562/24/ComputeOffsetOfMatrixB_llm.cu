#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize kernel for better memory coalescing and reduce redundant calculations
__global__ void ComputeOffsetOfMatrixB(const int32_t* row_sum, int32_t* output, int32_t N) {
    int rowStart = blockIdx.x * N;
    int offsetValue = -row_sum[blockIdx.x]; // Store common value for re-use

    for (int32_t i = threadIdx.x; i < N; i += blockDim.x) {
        *(output + rowStart + i) = offsetValue; // Use precomputed value
    }
}