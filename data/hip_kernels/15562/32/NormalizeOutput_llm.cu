#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize global kernel function
__global__ void NormalizeOutput(const int num_elements, const int* original, int64_t* to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (; idx < num_elements; idx += stride) { // Reuse stride variable in loop increment
        int base_idx = idx * 3; // Precompute base index
        to_normalize[base_idx] = batch_index;
        to_normalize[base_idx + 1] = class_index;
        to_normalize[base_idx + 2] = static_cast<int64_t>(original[idx]);
    }
}