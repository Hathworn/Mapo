#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Kernel to compute accuracy by dividing with N
__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Using block and thread index to potentially process multiple elements in parallel
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) { 
        accuracy[idx] /= N; // Perform division on each element
    }
}