#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GPU_increment_number(int* buffer, int initial)
{
    // Use threadIdx.x to handle multiple threads for scalability
    int idx = threadIdx.x;
    if (idx == 0) {
        buffer[0] = 1 + initial; // Efficiently handle the operation with the first thread
    }
}