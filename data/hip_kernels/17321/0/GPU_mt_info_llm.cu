#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

__global__ void GPU_mt_info()
{
    // Cache block and thread indices to reduce repeated accesses
    int block_index = blockIdx.x;
    int thread_index = threadIdx.x;

    printf("Block idx: %d | thread idx: %d\n", block_index, thread_index);
}