#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void GPU_vector_add(int* left, int* right, int* result, int size)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check to prevent out-of-bounds access
    if (idx < size) {
        result[idx] = left[idx] + right[idx];
    }
}