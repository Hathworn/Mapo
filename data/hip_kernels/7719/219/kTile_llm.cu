#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* src, float* tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    const uint numEls = tgtWidth * tgtHeight;

    for (uint i = idx; i < numEls; i += numThreads) {
        // Calculate y and x positions in the target
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;
        // Calculate corresponding positions in the source
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;
        // Map the value from source to target
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}