#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];
    
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f;

    // Loop unrolling for processing multiple elements per thread
    while (eidx < numElements) {
        sum += a[eidx] * b[eidx];
        eidx += gridDim.x * DP_BLOCKSIZE;
    }

    shmem[threadIdx.x] = sum;
    __syncthreads();
    
    // Reduce shared memory using warp shuffle
    if (threadIdx.x < 32) {
        volatile float* mysh = shmem;
        if (DP_BLOCKSIZE >= 64) mysh[threadIdx.x] += mysh[threadIdx.x + 32];
        if (DP_BLOCKSIZE >= 32) mysh[threadIdx.x] += mysh[threadIdx.x + 16];
        if (DP_BLOCKSIZE >= 16) mysh[threadIdx.x] += mysh[threadIdx.x + 8];
        if (DP_BLOCKSIZE >= 8) mysh[threadIdx.x] += mysh[threadIdx.x + 4];
        if (DP_BLOCKSIZE >= 4) mysh[threadIdx.x] += mysh[threadIdx.x + 2];
        if (DP_BLOCKSIZE >= 2) mysh[threadIdx.x] += mysh[threadIdx.x + 1];
    }

    // Write the result of the block to global memory
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}