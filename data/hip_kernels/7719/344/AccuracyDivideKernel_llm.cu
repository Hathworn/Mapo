#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load accuracy from global memory to a register for efficiency
    float acc = *accuracy;
    
    // Perform division
    acc /= N;
    
    // Store the result back to global memory
    *accuracy = acc;
}