#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate the unique thread index
    const uint tidx = blockDim.x * blockIdx.x + threadIdx.x;
    // Initialize the random state per thread
    hiprand_init(seed, tidx, 0, &state[tidx]);
}