#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kLogregCost(float* probs, float* labels, float* maxProbs, float* labelLogProbs, float* correctProbs, const int numCases, const int numOut) {
    const int tx = blockIdx.x * LOGREG_ERR_THREADS_X + threadIdx.x;

    // Ensure within bounds
    if (tx < numCases) {
        const int label = int(labels[tx]);
        const float maxp = maxProbs[tx];
        const float labelp = probs[label * numCases + tx];

        labelLogProbs[tx] = __logf(labelp);

        // Use shared memory for optimizations
        extern __shared__ int sharedCount[];
        sharedCount[threadIdx.x] = 0;

        // Compute the number of labels with max probability
        if (labelp == maxp) {
            for (int i = threadIdx.x; i < numOut; i += blockDim.x) {
                if (probs[i * numCases + tx] == maxp) {
                    atomicAdd(&sharedCount[threadIdx.x], 1);
                }
            }
        }
        __syncthreads();

        // Accumulate results in one thread
        if (threadIdx.x == 0) {
            int numMax = 0;
            for (int i = 0; i < blockDim.x; i++) {
                numMax += sharedCount[i];
            }
            if (labelp == maxp) {
                correctProbs[tx] = 1.0f / float(numMax);
            } else {
                correctProbs[tx] = 0;
            }
        }
    }
}