#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sigmoid64(double* A, int size)
{
    // Optimize block and thread index calculation
    int idx = blockIdx.x * blockDim.x + threadIdx.x 
            + blockIdx.y * (blockDim.x * gridDim.x) 
            + blockIdx.z * (blockDim.x * gridDim.x * gridDim.y);
    // Check and process within bounds
    if (idx < size) {
        A[idx] = 1.0 / (1.0 + exp(-A[idx])); // Use exp function for clarity and efficiency
    }
}