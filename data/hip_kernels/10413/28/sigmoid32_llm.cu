#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void sigmoid32(float* A, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified index calculation for 1D grid and block
    if (idx < size) {  // Adjusted condition for valid index range
        A[idx] = 1.0f / (1.0f + expf(-A[idx]));  // Direct call to expf for better performance
    }
}