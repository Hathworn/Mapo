#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    // Use shared memory for optimization
    extern __shared__ float shared_mem[];
    float *shared_input = shared_mem;

    // Load input into shared memory
    int tid = threadIdx.x;
    if (tid < n) {
        shared_input[tid] = input[tid * stride];
    }
    __syncthreads();

    float sum = 0;
    float largest = -INFINITY;

    // Use parallel reduction to find the largest value
    for (int offset = 1; offset < n; offset *= 2) {
        if (tid % (2 * offset) == 0 && (tid + offset) < n) {
            shared_input[tid] = max(shared_input[tid], shared_input[tid + offset]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        largest = shared_input[0];
    }
    __syncthreads();

    // Compute exponential values and sum
    if (tid < n) {
        float e = expf((shared_input[tid] - largest) / temp);
        shared_mem[tid] = e;
        atomicAdd(&sum, e);
    }
    __syncthreads();

    // Normalize the output
    if (tid < n) {
        output[tid * stride] = shared_mem[tid] / sum;
    }
}

__global__ void softmax_kernel(float *input, int n, int batch, int batch_offset, int groups, int group_offset, int stride, float temp, float *output)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= batch * groups) return;
    int b = id / groups;
    int g = id % groups;
    extern __shared__ float shared_mem[];
    int shared_mem_size = n * sizeof(float); // Adjust based on the 'n' size
    softmax_device(input + b*batch_offset + g*group_offset, n, temp, stride, output + b*batch_offset + g*group_offset, shared_mem, shared_mem_size);
}