#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_bias_kernel(float *output, float *biases, int n, int size)
{
    // Calculate indices for x, y, z dimensions
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Improved naming for readibility
    int filter = blockIdx.y;
    int batch = blockIdx.z;

    // Check boundary condition
    if (idx < size) {
        // Calculate the linear position in the output array
        int linear_idx = (batch * n + filter) * size + idx;
        // Multiply the output by the corresponding bias
        output[linear_idx] *= biases[filter];
    }
}