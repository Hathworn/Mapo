#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    // Calculate unique global thread ID
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Calculate multi-dimensional indices
    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -(size / 2);
    int h_offset = -(size / 2);
    int out_index = j + w * (i + h * (k + c * b));

    // Initialize accumulation variable
    float acc = 0.0f;

    // Optimize loop by combining condition check within processing
    for (int l = 0; l < size; ++l) {
        int cur_h = h_offset + i + l;
        bool h_valid = (cur_h >= 0 && cur_h < h);
        for (int m = 0; m < size; ++m) {
            int cur_w = w_offset + j + m;
            if (h_valid && cur_w >= 0 && cur_w < w) {
                int index = cur_w + w * (cur_h + h * (k + b * c));
                acc += rate * (x[index] - x[out_index]);
            }
        }
    }
    // Update delta after accumulation
    delta[out_index] += acc;
}