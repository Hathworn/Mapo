#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void pow_kernel(int N, float ALPHA, float *X, int INCX, float *Y, int INCY)
{
    // Utilizing grid-stride loop to handle more elements, improve memory access, and allow more flexibility on thread and block configuration
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < N; i += blockDim.x * gridDim.x) 
    {
        Y[i * INCY] = powf(X[i * INCX], ALPHA); // Use powf for single precision
    }
}