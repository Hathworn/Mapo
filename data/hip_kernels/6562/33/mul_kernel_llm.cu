#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mul_kernel(int N, float *X, int INCX, float *Y, int INCY)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplified block index computation
    int stride = gridDim.x * blockDim.x; // Calculate total number of threads
    for (; i < N; i += stride) { // Loop to cover full range
        Y[i * INCY] *= X[i * INCX];
    }
}