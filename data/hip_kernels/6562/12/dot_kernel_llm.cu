#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void dot_kernel(float *output, float scale, int batch, int n, int size, float *delta)
{
    // Calculate global index
    int index = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    int f1 = index / n;
    int f2 = index % n;
    
    // Early exit for invalid threads
    if (f2 <= f1) return;

    float sum = 0;
    float norm1 = 0;
    float norm2 = 0;
    
    // Unrolling batch loop for performance
    #pragma unroll
    for(int b = 0; b < batch; ++b){
        // Unrolling size loop
        #pragma unroll
        for(int i = 0; i < size; ++i){
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            sum += output[i1] * output[i2];
            norm1 += output[i1] * output[i1];
            norm2 += output[i2] * output[i2];
        }
    }
    
    // Compute norms and normalize sum
    norm1 = sqrt(norm1);
    norm2 = sqrt(norm2);
    float norm = norm1 * norm2;
    sum = sum / norm;

    // Update deltas with computed values
    #pragma unroll
    for(int b = 0; b < batch; ++b){
        #pragma unroll
        for(int i = 0; i < size; ++i){
            int i1 = b * size * n + f1 * size + i;
            int i2 = b * size * n + f2 * size + i;
            float update_val = -scale * sum / norm;
            delta[i1] += update_val * output[i2];
            delta[i2] += update_val * output[i1];
        }
    }
}