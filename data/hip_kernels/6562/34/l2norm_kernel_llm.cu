#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void l2norm_kernel(int N, float *x, float *dx, int batch, int filters, int spatial)
{
    // Calculate global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= N) return;
    
    // Calculate the batch and spatial index
    int b = index / spatial;
    int i = index % spatial;
    float sum = 0.0f;

    // Use shared memory for partial sum to reduce global memory accesses
    extern __shared__ float sdata[];
    for (int f = threadIdx.x; f < filters; f += blockDim.x) {
        int idx = b * filters * spatial + f * spatial + i;
        atomicAdd(&sdata[threadIdx.x], powf(x[idx], 2));
    }

    // Block-wide reduction for sum to calculate l2 norm
    __syncthreads();
    if (threadIdx.x == 0) {
        for (int j = 1; j < blockDim.x; j++) {
            sum += sdata[j];
        }
        sum = sqrtf(sum);
        if (sum == 0) sum = 1.0f;
    }
    __syncthreads();

    // Normalize x and compute dx using the l2 norm
    for (int f = threadIdx.x; f < filters; f += blockDim.x) {
        int idx = b * filters * spatial + f * spatial + i;
        x[idx] /= sum;
        dx[idx] = (1 - x[idx]) / sum;
    }
}