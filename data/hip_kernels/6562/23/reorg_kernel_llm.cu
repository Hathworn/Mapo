#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reorg_kernel(int N, float *x, int w, int h, int c, int batch, int stride, int forward, float *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Early exit if index is greater than N
    if(i >= N) return;

    // Calculate input indices
    int in_w = i % w;
    int in_h = (i / w) % h;
    int in_c = (i / w / h) % c;
    int b = i / w / h / c;

    // Calculate output indices
    int out_c = c / (stride * stride);
    int c2 = in_c % out_c;
    int offset = in_c / out_c;
    int w2 = in_w * stride + offset % stride;
    int h2 = in_h * stride + offset / stride;
    int out_index = w2 + w * stride * (h2 + h * stride * (c2 + out_c * b));

    // Conditional assignment based on forward flag
    if(forward) 
        out[out_index] = x[in_index];
    else 
        out[in_index] = x[out_index];
}