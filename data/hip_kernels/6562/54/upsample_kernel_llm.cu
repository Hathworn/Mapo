#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void upsample_kernel(size_t N, float *x, int w, int h, int c, int batch, int stride, int forward, float scale, float *out)
{
    // Calculate unique global index in the grid
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;

    // Early exit if index exceeds the range
    if(i >= N) return;

    // Use integer division and modulus in a single step
    int out_w = i % (w * stride);
    int out_h = (i / (w * stride)) % (h * stride);
    int out_c = (i / (w * stride) / (h * stride)) % c;
    int b = (i / (w * stride) / (h * stride) / c);

    int in_w = out_w / stride;
    int in_h = out_h / stride;
    int in_c = out_c;

    // Pre-calculate index for input array
    int in_index = b * w * h * c + in_c * w * h + in_h * w + in_w;

    // Efficiently handle forward and backward computation
    if(forward) 
        out[i] += scale * x[in_index];
    else 
        atomicAdd(x + in_index, scale * out[i]);
}