#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_kernel(float *bias_updates, float *delta, int batch, int n, int size)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    float sum = 0.0f;

    // Loop unrolling to enhance performance
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) {
            int index = i + size * (filter + n * b);
            sum += delta[index];
        }
    }
    part[p] = sum;
    __syncthreads();

    // Use parallel reduction to accumulate results
    for (int stride = BLOCK / 2; stride > 0; stride >>= 1) {
        if (p < stride) {
            part[p] += part[p + stride];
        }
        __syncthreads();
    }

    // Write back the final result to global memory
    if (p == 0) {
        bias_updates[filter] += part[0];
    }
}