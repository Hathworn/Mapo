#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void deinter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = blockDim.x * gridDim.x;  // Calculate gridSize to handle large data

    for (; i < (NX + NY) * B; i += gridSize) {  // Loop to process all data in chunks
        int b = i / (NX + NY);
        int j = i % (NX + NY);
        
        if (j < NX) {
            if(X) atomicAdd(&X[b*NX + j], OUT[i]);  // Use atomicAdd for concurrent writes
        } else {
            if(Y) atomicAdd(&Y[b*NY + j - NX], OUT[i]);  // Use atomicAdd for concurrent writes
        }
    }
}