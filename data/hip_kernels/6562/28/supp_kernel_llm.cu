#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void supp_kernel(int N, float ALPHA, float *X, int INCX)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Simplify block index calculation
    int stride = gridDim.x * blockDim.x; // Calculate the total number of threads

    for (; i < N; i += stride) { // Use a stride loop to cover all elements
        float val = X[i * INCX]; // Cache X[i*INCX] to reduce repeated computation
        if (val * val < ALPHA * ALPHA) 
            X[i * INCX] = 0;
    }
}