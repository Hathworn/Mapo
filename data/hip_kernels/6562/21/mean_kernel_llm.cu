#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    // Calculate scale outside the loop for efficiency
    float scale = 1.f / (batch * spatial);
    int i = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float temp_sum = 0.0f;  // Use a temporary variable to accumulate sum

    // Unroll the loop if possible, minor optimization
    int batch_spatial = batch * spatial;
    for (int j = 0; j < batch_spatial; ++j) {
        int index = j * filters + i;
        temp_sum += x[index];
    }
    
    mean[i] = temp_sum * scale;  // Apply scale at the end
}