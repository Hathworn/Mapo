#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Use variable to cache constant multiplication for better efficiency
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (i < N) {
        X[i * INCX] += ALPHA; // Minimize arithmetic operations
    }
}