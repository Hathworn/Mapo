#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_kernel(int N, float *x, float *mean, float *variance, int batch, int filters, int spatial)
{
    // Use threadIdx.x to calculate the index directly and add a branch-less approach for out-of-bound checks
    int index = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x; 
    if (index < N) {
        int f = (index / spatial) % filters; // Calculate filter index
        float var_adjust = sqrtf(variance[f] + .00001f); // Compute adjusted variance
        x[index] = (x[index] - mean[f]) / var_adjust; // Normalize
    }
}