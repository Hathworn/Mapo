#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void const_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Process multiple elements with a single thread (loop unrolling)
    for (; i < N; i += blockDim.x * gridDim.x)
    {
        X[i * INCX] = ALPHA;
    }
}