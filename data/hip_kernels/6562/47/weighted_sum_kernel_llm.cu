#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_sum_kernel(int n, float *a, float *b, float *s, float *c)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    // Unrolling to enhance performance if i < n
    if(i < n) {
        float s_val = s[i];
        float a_val = a[i];
        float b_val = (b ? b[i] : 0);
        
        // Pre-calculate constant terms and avoid accessing memory twice
        float weighted_a = s_val * a_val;
        float weighted_b = (1.0f - s_val) * b_val;

        c[i] = weighted_a + weighted_b;
    }
}