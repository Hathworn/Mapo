#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void flatten_kernel(int N, float *x, int spatial, int layers, int batch, int forward, float *out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;  // Calculate global thread index

    if(idx >= N) return;

    // Unroll the loop for better performance
    int b = idx / (layers * spatial);
    int rem = idx % (layers * spatial);
    int in_c = rem / spatial;
    int in_s = rem % spatial;

    int i1 = b * layers * spatial + in_c * spatial + in_s;
    int i2 = b * layers * spatial + in_s * layers + in_c;

    if (forward) 
        out[i2] = x[i1];
    else 
        out[i1] = x[i2];
}