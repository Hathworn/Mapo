#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_bias_conn_kernel(float *bias_updates, float *delta, int batch, int n)
{
    // Calculate the global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= n) return;

    // Use shared memory to reduce global memory access
    __shared__ float shared_sum[256]; // assuming blockDim.x <= 256
    shared_sum[threadIdx.x] = 0;

    // Accumulate sum for each batch
    for (int b = 0; b < batch; ++b) {
        int i = b * n + index;
        shared_sum[threadIdx.x] += delta[i];
    }

    // Perform reduction within block
    __syncthreads();
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            shared_sum[threadIdx.x] += shared_sum[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Add the partial sums to global memory
    if (threadIdx.x == 0) {
        atomicAdd(&bias_updates[index], shared_sum[0]);  // Use atomic operation to avoid race conditions
    }
}