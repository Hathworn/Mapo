#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void forward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *input, float *output, int *indexes)
{
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int c = in_c;

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= n) return;

    // Pre-calculate offsets for repeated ID index divisions
    int temp_id = id;
    int j = temp_id % w; temp_id /= w;
    int i = temp_id % h; temp_id /= h;
    int k = temp_id % c; temp_id /= c;
    int b = temp_id;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    int out_index = j + w * (i + h * (k + c * b));
    float max = -INFINITY;
    int max_i = -1;

    // Prefetch input dimensions to minimize register pressure within loop
    int input_w = in_w;
    int input_h = in_h;
    int input_c = in_c;

    // Use flattened loops for better warp utilization
    for (int l_m = 0; l_m < size * size; ++l_m) {
        int l = l_m / size;
        int m = l_m % size;
        int cur_h = h_offset + i * stride + l;
        int cur_w = w_offset + j * stride + m;
        int index = cur_w + input_w * (cur_h + input_h * (k + b * input_c));
        
        // Conditional check in one line for clarity and efficiency
        int valid = (cur_h >= 0 && cur_h < input_h && cur_w >= 0 && cur_w < input_w);
        
        float val = valid ? input[index] : -INFINITY;
        if (val > max) {
            max = val;
            max_i = index;
        }
    }
    output[out_index] = max;
    indexes[out_index] = max_i;
}