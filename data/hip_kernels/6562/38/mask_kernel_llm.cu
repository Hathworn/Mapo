#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mask_kernel(int n, float *x, float mask_num, float *mask, float val)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Unroll loop to improve memory access pattern
    for (int index = i; index < n; index += blockDim.x * gridDim.x)
    {
        // Use short-circuit evaluation for better performance
        if (mask[index] == mask_num)
        {
            x[index] = val;
        }
    }
}