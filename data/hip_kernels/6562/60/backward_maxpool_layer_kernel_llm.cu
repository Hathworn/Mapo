#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_maxpool_layer_kernel(int n, int in_h, int in_w, int in_c, int stride, int size, int pad, float *delta, float *prev_delta, int *indexes)
{
    int h = (in_h + pad - size) / stride + 1;
    int w = (in_w + pad - size) / stride + 1;
    int c = in_c;

    int id = blockIdx.x * blockDim.x + threadIdx.x;  // Optimized calculation of the unique thread ID
    if(id >= n) return;

    int index = id;
    int j = index % in_w;
    index /= in_w;
    int i = index % in_h;
    index /= in_h;
    int k = index % in_c;
    index /= in_c;
    int b = index;

    int w_offset = -pad / 2;
    int h_offset = -pad / 2;

    float d = 0.0f;
    int area = (size - 1) / stride;

    // Use single nested loop and optimize boundary checking
    for (int l = -area; l <= area; ++l) {
        for (int m = -area; m <= area; ++m) {
            int out_w = (j - w_offset) / stride + m;
            int out_h = (i - h_offset) / stride + l;
            if (out_w >= 0 && out_w < w && out_h >= 0 && out_h < h) {
                int out_index = out_w + w * (out_h + h * (k + c * b));
                d += (indexes[out_index] == id) ? delta[out_index] : 0.0f;  // Use id instead of index directly
            }
        }
    }

    // Avoid atomic operation by accumulating in local variable first
    prev_delta[id] += d;
}