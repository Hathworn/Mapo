#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void normalize_delta_kernel(int N, float *x, float *mean, float *variance, float *mean_delta, float *variance_delta, int batch, int filters, int spatial, float *delta)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Early return for invalid index
    if (index >= N) 
        return;
    
    int f = (index/spatial) % filters;
    float var_inv_sqrt = 1.f / sqrtf(variance[f] + .00001f); // Precompute inverse sqrt of variance
    float common_term = mean_delta[f] / (spatial * batch);  // Precompute common term

    // Optimize delta computation by reusing computations
    delta[index] = (delta[index] * var_inv_sqrt) + (variance_delta[f] * 2.f * (x[index] - mean[f]) / (spatial * batch)) + common_term;
}