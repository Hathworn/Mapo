#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void constrain_kernel(int N, float ALPHA, float *X, int INCX)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x; // Combine thread indexing
    int stride = gridDim.x * blockDim.x; // Calculate stride for grid stride loop
    for (int i = tid; i < N; i += stride) {  // Use grid stride loop for better parallelism
        X[i*INCX] = fminf(ALPHA, fmaxf(-ALPHA, X[i*INCX])); // Constrain value
    }
}