#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_scale_kernel(float *x_norm, float *delta, int batch, int n, int size, float *scale_updates)
{
    __shared__ float part[BLOCK];
    int filter = blockIdx.x;
    int p = threadIdx.x;
    
    float sum = 0;
    for (int b = 0; b < batch; ++b) {
        for (int i = p; i < size; i += BLOCK) {  // Optimize loop increment to use thread index
            int index = i + size * (filter + n * b);
            sum += delta[index] * x_norm[index];
        }
    }

    part[p] = sum;
    __syncthreads();

    // Use reduction to compute the sum across the shared memory
    if (p == 0) {
        float block_sum = 0;
        for (int i = 0; i < BLOCK; ++i) {
            block_sum += part[i];
        }
        atomicAdd(&scale_updates[filter], block_sum);  // Use atomicAdd for concurrency
    }
}