#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mult_add_into_kernel(int n, float *a, float *b, float *c)
{
    // Calculate the global index directly using 1D grid and block
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Use i < n check to avoid out-of-bounds access
    if (i < n) {
        c[i] += a[i] * b[i]; // Perform the multiplication and addition
    }
}