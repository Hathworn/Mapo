#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Each thread initializes its own local element to 0
    float local_sum = 0.0f;

    // Unroll outer loop for better performance
    for(int j = 0; j < batch; ++j){
        for(int i = id; i < spatial; i += threads){
            int index = j * spatial * filters + filter * spatial + i;
            local_sum += delta[index] * (x[index] - mean[filter]);
        }
    }

    // Store each thread's local sum to shared memory
    local[id] = local_sum;

    __syncthreads();

    // Perform reduction only if it's thread 0
    if(id == 0){
        float sum = 0.0f;
        for(int i = 0; i < threads; ++i){
            sum += local[i];
        }
        variance_delta[filter] = sum * -.5f * powf(variance[filter] + .00001f, -1.5f);
    }
}