#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void add_bias_kernel(float *output, float *biases, int batch, int n, int size)
{
    // Calculate the global index
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    // Use stride loop to allow blocks to handle more work
    int stride = blockDim.x * gridDim.x;

    for (int idx = index; idx < n * size * batch; idx += stride) {
        int i = idx % size;
        int j = (idx / size) % n;
        int k = idx / (n * size);

        output[(k * n + j) * size + i] += biases[j];
    }
}