#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void adam_kernel(int N, float *x, float *m, float *v, float B1, float B2, float rate, float eps, int t)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x; // Simplified index calculation
    if (index >= N) return;

    // Precompute powf(B1, t) and powf(B2, t) as they are the same for each thread
    float one_minus_pow_B1_t = 1.f - powf(B1, t);
    float one_minus_pow_B2_t = 1.f - powf(B2, t);
    float sqrt_eps = eps; // Precompute eps constant addition outside the loop

    // Leverage reuse of powf computation
    float mhat = m[index] / one_minus_pow_B1_t;
    float vhat = v[index] / one_minus_pow_B2_t;

    // Save computation by reusing variables
    x[index] = x[index] + rate * mhat / (sqrtf(vhat) + sqrt_eps);
}