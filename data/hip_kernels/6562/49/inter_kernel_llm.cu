#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void inter_kernel(int NX, float *X, int NY, float *Y, int B, float *OUT)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified block index calculation
    int gridStride = blockDim.x * gridDim.x;  // Calculate grid stride to allow more iterations per thread
    
    while (i < (NX + NY) * B) {  // Use grid-stride loop for handling multiple elements per thread
        int b = i / (NX + NY);
        int j = i % (NX + NY);
        if (j < NX) {
            OUT[i] = X[b * NX + j];
        } else {
            OUT[i] = Y[b * NY + j - NX];
        }
        i += gridStride;  // Move to the next element managed by this thread
    }
}