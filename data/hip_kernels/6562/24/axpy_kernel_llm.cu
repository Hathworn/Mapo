#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void axpy_kernel(int N, float ALPHA, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Use loop unrolling for accessing elements
    for (; i < N; i += blockDim.x * gridDim.x)
    {
        Y[OFFY + i * INCY] += ALPHA * X[OFFX + i * INCX];
    }
}