#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    int filter = blockIdx.x;

    // Initialize local memory only if necessary to reduce memory operations
    local[id] = 0.0f;

    int j;
    for (j = 0; j < batch; ++j) {
        int index_offset = j * spatial * filters + filter * spatial;
        for (int i = id; i < spatial; i += threads) { // Strided access to utilize all threads
            int index = index_offset + i;
            // Use fmaf (fused multiply-add) for potential performance improvement
            float diff = x[index] - mean[filter];
            local[id] += diff * diff;
        }
    }

    __syncthreads();

    // Use reduction to accumulate values more efficiently
    for (int stride = threads / 2; stride > 0; stride /= 2) {
        if (id < stride) {
            local[id] += local[id + stride];
        }
        __syncthreads();
    }

    // Final calculation by the first thread in the block
    if (id == 0) {
        variance[filter] = local[0] / (spatial * batch - 1);
    }
}