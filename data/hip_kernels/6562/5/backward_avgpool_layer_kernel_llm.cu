#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void backward_avgpool_layer_kernel(int n, int w, int h, int c, float *in_delta, float *out_delta)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Calculate unique thread index
    if (id >= n) return;

    int k = id % c;
    int b = id / c; // Use integer division to compute b

    float scale = 1.0f / (w * h); // Precompute scale factor for average
    int out_index = k + c * b; 

    for (int i = 0; i < w * h; ++i)
    {
        int in_index = i + h * w * (k + b * c);
        in_delta[in_index] += out_delta[out_index] * scale; // Utilize precomputed scale
    }
}