#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory for partial reduction and avoid conditional operations
__global__ void l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;  // Simplified block index calculation
    if(i < n) {
        float diff = truth[i] - pred[i];
        error[i] = fabsf(diff);  // Use faster device function for absolute value
        delta[i] = copysignf(1.0f, diff);  // Use copysignf to simplify sign determination
    }
}