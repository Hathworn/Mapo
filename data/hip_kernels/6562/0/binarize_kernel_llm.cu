#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    // Calculate global thread ID
    int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Ensure the thread is within array bounds
    if (i < n) {
        // Use ternary operator for binarization
        binary[i] = (x[i] >= 0.0f) ? 1.0f : -1.0f;
    }
}