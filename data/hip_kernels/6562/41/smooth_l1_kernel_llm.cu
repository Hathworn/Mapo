#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void smooth_l1_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Ensure thread operates within bounds
    if(i < n) {
        // Calculate the difference between truth and prediction
        float diff = truth[i] - pred[i];
        float abs_val = fabsf(diff);
        
        // Use intrinsic operations for better performance
        if(abs_val < 1.0f) {
            error[i] = diff * diff;
            delta[i] = diff;
        }
        else {
            error[i] = fmaf(2.0f, abs_val, -1.0f);  // Use fmaf for fused multiply-add
            delta[i] = copysignf(1.0f, diff);     // Use copysignf for better readability
        }
    }
}