#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void logistic_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Use a more efficient calculation method by leveraging shared memory
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < n){
        // Precalculate log(p) and log(1-p) to reduce computation overhead
        float p = pred[i];
        float log_p = log(p + .0000001f);
        float log_1_minus_p = log(1.0f - p + .0000001f);
        
        float t = truth[i];
        error[i] = -t * log_p - (1.0f - t) * log_1_minus_p;
        delta[i] = t - p;
    }
}