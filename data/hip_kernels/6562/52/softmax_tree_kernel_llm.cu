#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void softmax_device(float *input, int n, float temp, int stride, float *output)
{
    float sum = 0.0f;
    float largest = -INFINITY;
    // Find the largest value
    for(int i = 0; i < n; ++i){
        float val = input[i * stride];
        largest = fmaxf(val, largest);
    }
    // Compute exponentials and sum
    for(int i = 0; i < n; ++i){
        float e = expf((input[i * stride] - largest) / temp);
        sum += e;
        output[i * stride] = e;
    }
    // Normalize output
    for(int i = 0; i < n; ++i){
        output[i * stride] /= sum;
    }
}

__global__ void softmax_tree_kernel(float *input, int spatial, int batch, int stride, float temp, float *output, int groups, int *group_size, int *group_offset)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= spatial * batch * groups) return;

    int s = id % spatial;
    id /= spatial;
    int g = id % groups;
    int b = id / groups;

    int goff = group_offset[g] * spatial;
    int boff = b * stride;

    // Call to the optimized device softmax function
    softmax_device(input + goff + boff + s, group_size[g], temp, spatial, output + goff + boff + s);
}