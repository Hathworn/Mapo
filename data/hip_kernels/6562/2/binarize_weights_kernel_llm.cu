#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;

    // Use shared memory to reduce global memory access latency
    extern __shared__ float shared_weights[];
    
    float mean = 0;
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[threadIdx.x] = fabsf(weights[f*size + i]);
        __syncthreads(); 
        mean += shared_weights[threadIdx.x];
        __syncthreads();
    }
    
    // Calculate mean in parallel
    mean = mean / size;
    
    for (int i = threadIdx.x; i < size; i += blockDim.x) {
        shared_weights[threadIdx.x] = weights[f*size + i];
        __syncthreads();
        binary[f*size + i] = (shared_weights[threadIdx.x] > 0) ? mean : -mean;
        __syncthreads();
    }
}