#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize HSV to RGB conversion kernel
__device__ float3 hsv_to_rgb_kernel(float3 hsv)
{
    const float h = hsv.x, s = hsv.y, v = hsv.z;
    if (s == 0) return make_float3(v, v, v);
    
    const int index = (int)floorf(h);
    const float f = h - index;
    const float p = v * (1 - s);
    const float q = v * (1 - s * f);
    const float t = v * (1 - s * (1 - f));

    if (index == 0) return make_float3(v, t, p);
    if (index == 1) return make_float3(q, v, p);
    if (index == 2) return make_float3(p, v, t);
    if (index == 3) return make_float3(p, q, v);
    if (index == 4) return make_float3(t, p, v);
    return make_float3(v, p, q);
}

// Optimize RGB to HSV conversion kernel
__device__ float3 rgb_to_hsv_kernel(float3 rgb)
{
    const float r = rgb.x, g = rgb.y, b = rgb.z;
    const float max = fmaxf(fmaxf(r, g), b);
    const float min = fminf(fminf(r, g), b);
    const float delta = max - min;
    float h = -1, s = max == 0 ? 0 : delta / max, v = max;

    if (delta != 0) {
        if (r == max) h = (g - b) / delta;
        else if (g == max) h = 2 + (b - r) / delta;
        else h = 4 + (r - g) / delta;
        if (h < 0) h += 6;
    }
    return make_float3(h, s, v);
}

// Optimize global kernel function
__global__ void levels_image_kernel(float *image, float *rand, int batch, int w, int h, int train, float saturation, float exposure, float translate, float scale, float shift)
{
    int size = batch * w * h;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id >= size) return;
   
    float rshift = rand[0], gshift = rand[1], bshift = rand[2];
    float r0 = rand[8 * id + 0], r1 = rand[8 * id + 1], r2 = rand[8 * id + 2], r3 = rand[8 * id + 3];
    
    saturation = r0 * (saturation - 1) + 1;
    saturation = (r1 > .5f) ? 1.f / saturation : saturation;
    exposure = r2 * (exposure - 1) + 1;
    exposure = (r3 > .5f) ? 1.f / exposure : exposure;

    size_t offset = id * h * w * 3;
    float r = image[offset + w * (0 * h)], g = image[offset + w * (1 * h)], b = image[offset + w * (2 * h)];
    float3 rgb = make_float3(r, g, b);

    if (train) {
        float3 hsv = rgb_to_hsv_kernel(rgb);
        hsv.y *= saturation;
        hsv.z *= exposure;
        rgb = hsv_to_rgb_kernel(hsv);
    } else {
        shift = 0;
    }

    image[offset + w * (0 * h)] = rgb.x * scale + translate + (rshift - .5f) * shift;
    image[offset + w * (1 * h)] = rgb.y * scale + translate + (gshift - .5f) * shift;
    image[offset + w * (2 * h)] = rgb.z * scale + translate + (bshift - .5f) * shift;
}