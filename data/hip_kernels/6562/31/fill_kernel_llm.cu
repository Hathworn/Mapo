#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate a unique global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = gridDim.x * blockDim.x;

    // Loop to handle large arrays, more efficient than single if check
    for (; i < N; i += stride) {
        X[i * INCX] = ALPHA;
    }
}