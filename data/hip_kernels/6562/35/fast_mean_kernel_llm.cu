#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_kernel(float *x, int batch, int filters, int spatial, float *mean)
{
    const int threads = BLOCK;
    __shared__ float local[threads];
    
    int id = threadIdx.x;
    local[id] = 0;
    
    int filter = blockIdx.x;
    int stride = batch * spatial * filters;
    
    // Iterate over the batch spatial blocks in parallel
    for(int j = 0; j < batch; ++j) {
        int base_index = j * stride + filter * spatial;
        for(int i = threadIdx.x; i < spatial; i += threads) {
            int index = base_index + i;
            local[id] += x[index];
        }
    }
    
    __syncthreads();
    
    // Efficient reduction to compute mean
    if(id == 0) {
        float sum = 0;
        for(int i = 0; i < threads; ++i) {
            sum += local[i];
        }
        mean[filter] = sum / (spatial * batch);
    }
}