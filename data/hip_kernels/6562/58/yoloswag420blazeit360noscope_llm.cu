#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void yoloswag420blazeit360noscope(float *input, int size, float *rand, float prob, float scale)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; // Flatten 1D block index
    if (id < size) {
        float rand_value = rand[id];
        float input_value = input[id];
        // Conditional assignment with ternary operator
        input[id] = rand_value < prob ? 0.0f : input_value * scale;
    }
}