#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void forward_avgpool_layer_kernel(int n, int w, int h, int c, float *input, float *output)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int batch_size = n / c;  // Compute batch size
    int b = id / c;          // Calculate batch index
    int k = id % c;          // Calculate channel index

    float sum = 0;
    int start_index = h * w * (k + b * c); // Pre-compute start index
    for(int i = 0; i < w * h; ++i) {
        sum += input[start_index + i];
    }
    output[k + c * b] = sum / (w * h);      // Assign computed average to output
}