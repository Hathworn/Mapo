#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void weighted_delta_kernel(int n, float *a, float *b, float *s, float *da, float *db, float *ds, float *dc)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Flattened grid index for 1D block
    int stride = gridDim.x * blockDim.x; // Stride for grid-stride loop

    for (; i < n; i += stride) { // Grid-stride loop for improved memory coalescing
        float dc_val = dc[i];
        float s_val = s[i];
        if (da) da[i] += dc_val * s_val; // Use temporary variables
        if (db) db[i] += dc_val * (1.0f - s_val);
        ds[i] += dc_val * (a[i] - b[i]);
    }
}