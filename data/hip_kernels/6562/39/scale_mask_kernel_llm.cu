#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scale_mask_kernel(int n, float *x, float mask_num, float *mask, float scale)
{
    // Calculate global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridStride = gridDim.x * blockDim.x;

    // Loop through data in a grid-stride loop for better workload distribution
    for (; i < n; i += gridStride) {
        if (mask[i] == mask_num) {
            x[i] *= scale;
        }
    }
}