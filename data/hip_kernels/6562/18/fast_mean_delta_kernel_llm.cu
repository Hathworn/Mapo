#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fast_mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    const int threads = BLOCK;
    __shared__ float local[threads];

    int id = threadIdx.x;
    local[id] = 0;

    int filter = blockIdx.x;

    int i, j;
    for(j = 0; j < batch; ++j){
        for(i = id; i < spatial; i += threads){
            int index = j*spatial*filters + filter*spatial + i;
            local[id] += delta[index];
        }
    }

    __syncthreads();

    // Using reduction technique to accumulate results
    if (id < threads / 2) {
        local[id] += local[id + threads / 2];
    }
    __syncthreads();

    if (id < threads / 4) {
        local[id] += local[id + threads / 4];
    }
    __syncthreads();

    if (id < threads / 8) {
        local[id] += local[id + threads / 8];
    }
    __syncthreads();

    if (id < threads / 16) {
        local[id] += local[id + threads / 16];
    }
    __syncthreads();

    if (id == 0) {
        // Final sum computation by the first thread
        mean_delta[filter] = local[0] * (-1.f / sqrtf(variance[filter] + .00001f));
    }
}