#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void l2_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Calculate unique thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Ensure the thread is within the bounds
    if (i < n) {
        float diff = truth[i] - pred[i];
        error[i] = diff * diff;  // Calculate squared error
        delta[i] = diff;         // Store the difference
    }
}