#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mean_delta_kernel(float *delta, float *variance, int batch, int filters, int spatial, float *mean_delta)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;
    
    float sum = 0; // Use a local variable for accumulation
    int base_idx = i * spatial; // Compute base indexing outside of inner loop
    
    for (int j = 0; j < batch; ++j) {
        int batch_start = j * filters * spatial; // Precompute start index for batch
        for (int k = 0; k < spatial; ++k) {
            sum += delta[batch_start + base_idx + k];
        }
    }
    
    mean_delta[i] = sum * (-1.f / sqrtf(variance[i] + .00001f)); // Compute result outside of loops
}