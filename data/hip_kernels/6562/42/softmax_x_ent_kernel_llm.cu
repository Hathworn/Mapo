#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void softmax_x_ent_kernel(int n, float *pred, float *truth, float *delta, float *error)
{
    // Optimize by using a more precise calculation for global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = gridDim.x * blockDim.x; // Calculate total threads

    while (i < n) { // Loop to handle more elements per thread
        float t = truth[i];
        float p = pred[i];
        error[i] = (t > 0.0f) ? -__logf(p) : 0.0f; // Use __logf for faster computation
        delta[i] = t - p;
        i += gridSize; // Advance by total number of threads to process next element
    }
}