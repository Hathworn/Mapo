#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    // Using combined index in one statement for potential compiler optimization
    int s = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;
    if (s >= size) return;

    // Initializing mean calculation variables
    float sum = 0.0f;

    // Calculate mean value in a more GPU-efficient way by using registers
    for (int i = 0; i < n; ++i) {
        sum += fabsf(input[i * size + s]);
    }
    float mean = sum / n;

    // Write to output in a coalesced manner
    for (int i = 0; i < n; ++i) {
        binary[i * size + s] = (input[i * size + s] > 0) ? mean : -mean;
    }
}