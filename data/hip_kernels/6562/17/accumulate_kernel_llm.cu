#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void accumulate_kernel(float *x, int n, int groups, float *sum)
{
    // Calculate global thread index for better scalability
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Check if thread index is within bounds
    if (i >= groups) return;

    // Use shared memory for partial sums to reduce global memory access
    extern __shared__ float shared_sum[];
    shared_sum[threadIdx.x] = 0;

    // Optimize loop by unrolling for better performance
    for (int k = 0; k < n; ++k){
        shared_sum[threadIdx.x] += x[k * groups + i];
    }
    
    // Write the computed sum back to global memory
    sum[i] = shared_sum[threadIdx.x];
}