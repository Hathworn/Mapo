#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void variance_delta_kernel(float *x, float *delta, float *mean, float *variance, int batch, int filters, int spatial, float *variance_delta)
{
    // Calculate unique thread index
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;

    // Exit if thread is out of bounds
    if (i >= filters) return;

    // Declare accumulator
    float acc = 0.0f;

    // Precompute multiplier for efficiency
    float variance_term = -.5f * powf(variance[i] + .00001f, (float)(-3.f/2.f));

    // Loop over batch and spatial dimensions with precomputed stride
    for (int j = 0; j < batch; ++j) {
        for (int k = 0; k < spatial; ++k) {
            // Compute linear index
            int index = j*filters*spatial + i*spatial + k;
            
            // Accumulate result
            acc += delta[index] * (x[index] - mean[i]);
        }
    }

    // Apply precomputed term and store result efficiently
    variance_delta[i] = acc * variance_term;
}