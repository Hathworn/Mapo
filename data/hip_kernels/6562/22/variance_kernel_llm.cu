#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void variance_kernel(float *x, float *mean, int batch, int filters, int spatial, float *variance)
{
    float scale = 1.f / (batch * spatial - 1);
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= filters) return;

    float var_accum = 0; // Use a local accumulator to reduce global memory access
    for(int j = 0; j < batch; ++j){
        for(int k = 0; k < spatial; ++k){
            int index = j * filters * spatial + i * spatial + k;
            float diff = x[index] - mean[i];
            var_accum += diff * diff; // Use multiplication instead of powf for efficiency
        }
    }
    variance[i] = var_accum * scale;
}