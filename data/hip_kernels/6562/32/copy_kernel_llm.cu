#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void copy_kernel(int N, float *X, int OFFX, int INCX, float *Y, int OFFY, int INCY)
{
    // Calculate the global index using built-in helper functions for clarity
    int i = hipBlockIdx_x * hipBlockDim_x + hipThreadIdx_x;

    // Check within bounds and perform copy operation
    if(i < N) {
        Y[i * INCY + OFFY] = X[i * INCX + OFFX];
    }
}