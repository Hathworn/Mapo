#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scal_kernel(int N, float ALPHA, float *X, int INCX)
{
    // Calculate global index to ensure thread safety
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Loop over stride to improve utilization
    for (; i < N; i += blockDim.x * gridDim.x) {
        X[i * INCX] *= ALPHA;
    }
}