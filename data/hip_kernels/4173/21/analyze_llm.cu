#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void analyze(const float *input, float *sum, int numElements) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Use shared memory for reducing frequent global memory access
    __shared__ float cache[256]; // assuming blockDim.x = 256
    if (i < numElements) {
        cache[threadIdx.x] = input[i];
    } else {
        cache[threadIdx.x] = 0.0f; // Handle out-of-bounds threads
    }
    __syncthreads();

    // Perform reduction within the block
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x < stride) {
            cache[threadIdx.x] += cache[threadIdx.x + stride];
        }
        __syncthreads();
    }

    // Write the result of this block to global memory
    if (threadIdx.x == 0) {
        atomicAdd(sum, cache[0]);
    }
}