#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(const float *input, float *output, int numElements) {
    // Get global thread index
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // Check if thread is within bounds
    if (tid < numElements) {
        // Use shared memory to accumulate results
        extern __shared__ float sharedData[];
        
        // Initialize shared memory
        sharedData[threadIdx.x] = input[tid];
        __syncthreads();

        // Perform reduction in shared memory
        for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
            if (threadIdx.x < stride) {
                sharedData[threadIdx.x] += sharedData[threadIdx.x + stride];
            }
            __syncthreads();
        }

        // Write result for this block to global memory
        if (threadIdx.x == 0) {
            atomicAdd(output, sharedData[0]);
        }
    }
}