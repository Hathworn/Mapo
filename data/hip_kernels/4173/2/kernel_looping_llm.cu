#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_looping(float *point, unsigned int num) {
    unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int stride = gridDim.x * blockDim.x;  // Pre-compute stride

    for (int iloop = 0; iloop < NLOOPS; ++iloop) {
        for (size_t offset = idx; offset < num; offset += stride) {
            point[offset] += 1;
        }
    }
}