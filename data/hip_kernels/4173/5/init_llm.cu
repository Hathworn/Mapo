#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(uint32_t *v, uint32_t val, uint32_t n) {
    auto i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        v[i] = val; // Efficiently assign value if within bounds
    }
    if (i == 0) {
        printf("init\n"); // Only one thread prints
    }
}