#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void addKernel(int *c, const int *a, const int *b)
{
    // Calculate the global index
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    
    // Perform addition if the index is within bounds
    if (i < N) // Assume N is the length of the arrays, defined elsewhere
    {
        c[i] = a[i] + b[i];
    }
}