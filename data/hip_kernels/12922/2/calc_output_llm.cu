#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc_output(unsigned char * img_out, unsigned char * img_in, int * lut, int img_size) {
    // Calculate the global thread index
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    const int gridW = gridDim.x * blockDim.x;
    int img_position1 = iy * gridW + ix;

    // Process only valid image positions
    if (img_position1 < img_size) {
        int lut_value = lut[img_in[img_position1]];
        // Use conditional operator for compactness
        img_out[img_position1] = (unsigned char)(lut_value > 255 ? 255 : lut_value);
    }
}