#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolutionRowGPU(double *h_Dst, double *h_Src, double *h_Filter, int imageW, int imageH, int filterR) {
    // Calculate global thread coordinates
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix < imageW && iy < imageH) { // Ensure we are within the bounds of the image
        double sum = 0;
        int k;
        // Perform convolution
        for (k = -filterR; k <= filterR; k++) {
            int d = ix + k;
            // Check if the filter position is within image bounds
            if (d >= 0 && d < imageW) {
                sum += h_Src[iy * imageW + d] * h_Filter[filterR - k];
            }
        }
        // Write the result to the destination array
        h_Dst[iy * imageW + ix] = sum;
    }
}