#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void convolutionColumnGPU(double *h_Dst, double *h_Src, double *h_Filter, int imageW, int imageH, int filterR) {
    // Calculate global row and column indices
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    if (ix >= imageW || iy >= imageH) return; // Boundary check

    double sum = 0;
    
    // Loop with optimized memory access
    for (int k = -filterR; k <= filterR; k++) {
        int d = iy + k;
        if (d >= 0 && d < imageH) {
            sum += h_Src[d * imageW + ix] * h_Filter[filterR - k];
        }
    }
    
    h_Dst[iy * imageW + ix] = sum; // Write result to global memory after the loop
}