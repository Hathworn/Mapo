#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Global histogram for all colors
__device__ int glb_hist[COLORS];

// Kernel for computing histogram between offsets
__global__ void calc_histogram(unsigned char *img_in, int offset_start, int offset_end) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int gridW = gridDim.x * blockDim.x;
    int pos;

    // Declare shared memory for block histograms
    __shared__ int hist[COLORS];

    // Initialize shared histogram to zero
    if (threadIdx.x < COLORS) {
        hist[threadIdx.x] = 0;
    }
    __syncthreads();

    // Calculate number of rows each thread will process
    int total_pixels = offset_end - offset_start;
    int pixels_per_thread = total_pixels / gridW;
    int start_pixel = ix * pixels_per_thread;
    int end_pixel = start_pixel + pixels_per_thread;
    
    // Process pixels assigned to this thread
    for (int i = start_pixel; i < end_pixel; ++i) {
        atomicAdd(&hist[img_in[i + offset_start]], 1);
    }

    // Handle remaining pixels
    if (ix < total_pixels % gridW) {
        pos = pixels_per_thread * gridW + ix;
        atomicAdd(&hist[img_in[pos + offset_start]], 1);
    }
    __syncthreads();

    // Update global histogram with block results
    if (threadIdx.x < COLORS) {
        atomicAdd(&glb_hist[threadIdx.x], hist[threadIdx.x]);
    }
}

// Kernel for computing histogram of an image
__global__ void calc_histogram(int *hist_out, unsigned char *img_in, int img_size, int nbr_bin) {
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    const int gridW = gridDim.x * blockDim.x;
    int img_position = iy * gridW + ix;

    // Initialize histogram output to zero
    if (img_position < nbr_bin) {
        hist_out[img_position] = 0;
    }
    __syncthreads();

    // Accumulate histogram for each pixel
    if (img_position < img_size) {
        atomicAdd(&hist_out[img_in[img_position]], 1);
    }
    __syncthreads();
}