#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int glb_hist[COLORS];

__global__ void calc_histogram(unsigned char * img_in, int offset_start, int offset_end){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    const int gridW = gridDim.x * blockDim.x;

    __shared__ int hist[COLORS];

    if (threadIdx.x < COLORS) {
        hist[threadIdx.x] = 0; // Initialize shared histogram
    }
    __syncthreads();

    int num_iterations = (offset_end - offset_start + gridW - 1) / gridW; // Calculate iterations handling leftovers
    for (int iter = 0; iter < num_iterations; iter++) {
        int pos = iter * gridW + ix;
        if (pos < (offset_end - offset_start)) {
            atomicAdd(&hist[img_in[pos + offset_start]], 1); // Accumulate local histogram
        }
    }
    __syncthreads();

    if (threadIdx.x < COLORS) {
        atomicAdd(&glb_hist[threadIdx.x], hist[threadIdx.x]); // Merge local histogram to global
    }
}

__global__ void calc_histogram(int * hist_out, unsigned char * img_in, int img_size, int nbr_bin){
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    const int gridW = gridDim.x * blockDim.x;
    int img_position = iy * gridW + ix; // Calculate image position

    if (img_position < nbr_bin) {
        hist_out[img_position] = 0; // Initialize histogram output
    }
    __syncthreads();

    if(img_position < img_size){
        atomicAdd(&hist_out[img_in[img_position]], 1); // Accumulate histogram
    }
    __syncthreads();
}