#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc_lut(int *lut, int *hist_in, int img_size, int nbr_bin) {
    __shared__ int shared_hist[256];
    __shared__ int cdf[256];

    // Load data into shared memory
    shared_hist[threadIdx.x] = hist_in[threadIdx.x];
    __syncthreads();

    // Calculate minimum non-zero value
    int i = 0, min = 0, d;
    while (min == 0 && i < 256) {
        min = shared_hist[i++];
    }
    d = img_size - min;

    // Prefix sum (CDF calculation)
    for (unsigned int stride = 1; stride < 256; stride *= 2) {
        if (threadIdx.x >= stride) {
            shared_hist[threadIdx.x] += shared_hist[threadIdx.x - stride];
        }
        __syncthreads();
    }

    cdf[threadIdx.x] = shared_hist[threadIdx.x];
    __syncthreads();

    // Compute LUT value
    lut[threadIdx.x] = (int)(((float)cdf[threadIdx.x] - min) * (nbr_bin - 1) / d + 0.5);
    if (lut[threadIdx.x] < 0) {
        lut[threadIdx.x] = 0;
    }
}