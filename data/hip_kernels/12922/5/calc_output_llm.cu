#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc_output(unsigned char * img_out, unsigned char * img_in, int * lut, int img_size) {
    // Calculate pixel index in the image
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int img_position = iy * (gridDim.x * blockDim.x) + ix;

    // Check if the pixel index is within image bounds
    if (img_position < img_size) {
        // Clamp LUT value to [0, 255] and assign to output image
        img_out[img_position] = min(255, lut[img_in[img_position]]);
    }
}