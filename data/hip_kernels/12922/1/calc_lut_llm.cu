#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void calc_lut(int *lut, int *hist_in, int img_size, int nbr_bin) {

    __shared__ int shared_hist[256];
    shared_hist[threadIdx.x] = hist_in[threadIdx.x];
    __syncthreads();

    int cdf = 0;
    int min = 0;
    int d;

    // Use atomic operation to find the minimum
    for (int i = 0; i < blockDim.x; i++) {
        atomicMin(&min, shared_hist[i]);
    }
    __syncthreads();

    if (min == 0) {
        return;  // Early exit if min is zero
    }

    d = img_size - min;

    // Compute prefix sum (CDF)
    for (int i = 0; i <= threadIdx.x; i++) {
        cdf += shared_hist[i];
    }

    // Calculate LUT value
    lut[threadIdx.x] = (int)(((float)cdf - min) * (nbr_bin - 1) / d + 0.5);

    // Ensure LUT value does not fall below zero
    if (lut[threadIdx.x] < 0) {
        lut[threadIdx.x] = 0;
    }
}