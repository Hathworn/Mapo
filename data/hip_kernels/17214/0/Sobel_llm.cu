#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#pragma diag_suppress integer_sign_change

static unsigned short* d_in;
static unsigned char* d_out;

static unsigned int h_Width;
static unsigned int h_Height;

static unsigned int h_BlockWidth;
static unsigned int h_BlockHeight;

#define THREAD_TOTAL_X_LEN 12
#define THREAD_AUX_X_LEN 4
#define THREAD_WORKING_X_LEN (THREAD_TOTAL_X_LEN - THREAD_AUX_X_LEN)

#define THREAD_TOTAL_Y_LEN 12
#define THREAD_AUX_Y_LEN 4
#define THREAD_WORKING_Y_LEN (THREAD_TOTAL_Y_LEN - THREAD_AUX_Y_LEN)

#define OFFSET(x,y) sIdx + y * THREAD_TOTAL_X_LEN + x

__global__ void Sobel(const unsigned short* in, unsigned char* out, const unsigned int width, const unsigned int height)
{
    extern __shared__ float s[];

    const unsigned int xPos = (blockIdx.x * THREAD_WORKING_X_LEN + threadIdx.x) - (THREAD_AUX_X_LEN / 2);
    const unsigned int yPos = (blockIdx.y * THREAD_WORKING_Y_LEN + threadIdx.y) - (THREAD_AUX_Y_LEN / 2);
    const unsigned int inPos = (xPos + yPos * width);
    const unsigned int sIdx = (threadIdx.x + threadIdx.y * THREAD_TOTAL_X_LEN);
    unsigned int outIt = inPos * 4;

    // Efficient shared memory usage
    if (xPos < width && yPos < height)
        s[sIdx] = in[inPos] / float(USHRT_MAX);
    else
        s[sIdx] = 0.0f;

    __syncthreads();

    // Optimize bounds checking within kernel
    if ((threadIdx.x - (THREAD_AUX_X_LEN / 2)) < THREAD_WORKING_X_LEN && (threadIdx.y - (THREAD_AUX_Y_LEN / 2)) < THREAD_WORKING_Y_LEN)
    {
        // Optimize Sobel calculations by clearly defining the sobel kernels
        const float sobelXKernel[5] = {-1, -2, 0, 2, 1};
        const float sobelYKernel[5] = {1, 2, 0, -2, -1};

        float sobelX = 0.0f;
        float sobelY = 0.0f;

        for (int i = -2; i <= 2; ++i) {
            sobelX += sobelXKernel[i + 2] * (s[OFFSET(i, -2)] + s[OFFSET(i, -1)] * 2 + s[OFFSET(i, 0)] * 4 + s[OFFSET(i, 1)] * 2 + s[OFFSET(i, 2)]);
            sobelY += sobelYKernel[i + 2] * (s[OFFSET(-2, i)] + s[OFFSET(-1, i)] * 2 + s[OFFSET(0, i)] * 4 + s[OFFSET(1, i)] * 2 + s[OFFSET(2, i)]);
        }
        sobelX *= (512 / 8) / 18;
        sobelY *= (512 / 8) / 18;

        const float gradientLen = sqrt(sobelX * sobelX + sobelY * sobelY + 1.0f);

        const unsigned char xLen = -(sobelX * 128) / gradientLen + 128;
        const unsigned char yLen = -(sobelY * 128) / gradientLen + 128;
        const unsigned char zLen = UCHAR_MAX / gradientLen;

        out[outIt++] = xLen;
        out[outIt++] = yLen;
        out[outIt++] = zLen;
        out[outIt] = 255;
    }
}