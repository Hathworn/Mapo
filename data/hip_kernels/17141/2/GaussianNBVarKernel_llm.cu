#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
    return (row * width + col);
}

__global__ void GaussianNBVarKernel(const float *d_data, const int *d_labels, const float *feature_means_, 
                                    float *feature_vars_, const int *class_count_, 
                                    const unsigned int n_samples_, const unsigned int n_classes_, 
                                    const unsigned int n_features_) {
    unsigned int tidx = threadIdx.x;
    unsigned int feat_col = tidx + (blockIdx.x * blockDim.x);

    if (feat_col < n_features_) {
        // Calculate variances using shared memory for faster access
        __shared__ float temp_vars[MAX_N_CLASSES];
        for (unsigned int i = 0; i < n_classes_; ++i) {
            temp_vars[i] = 0.0f; // Initialize shared memory
        }
        __syncthreads();

        for (unsigned int i = 0; i < n_samples_; ++i) {
            unsigned int row = d_labels[i];
            float diff = d_data[RM_Index(i, feat_col, n_features_)] - feature_means_[RM_Index(row, feat_col, n_features_)];
            atomicAdd(&temp_vars[row], diff * diff); // Use atomicAdd to avoid race condition
        }
        __syncthreads();

        for (unsigned int i = 0; i < n_classes_; ++i) {
            feature_vars_[RM_Index(i, feat_col, n_features_)] = temp_vars[i] / class_count_[i]; // Normalize using class count
        }
    }
}
```

Make sure to adjust `MAX_N_CLASSES` to correspond to the maximum expected number of classes that the shared memory can handle.