#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
    return (row * width + col);
}
__global__ void MultinomialNBLearnKernel(float *feature_probs, float *class_priors, const float *d_row_sums, unsigned int n_samples_, unsigned int n_classes_, unsigned int n_features_) {
    // Calculate unique global thread index
    unsigned int feat_col = blockIdx.x * blockDim.x + threadIdx.x;

    if (feat_col < n_features_) {
        for (unsigned int i = 0; i < n_classes_; ++i) {
            // Precompute common index calculation
            unsigned int idx = RM_Index(i, feat_col, n_features_);
            feature_probs[idx] = log((feature_probs[idx] + 1.0) / (d_row_sums[i] - n_features_));

            if (feat_col == 0) {
                // Compute and store class prior only once per class
                class_priors[i] = log(class_priors[i] / (float)n_samples_);
            }
        }
    }
}