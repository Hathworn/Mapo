#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
    return (row * width + col);
}

__global__ void BernoulliNBLearnKernel(float *feature_probs, float *class_count_, const float *d_row_sums, unsigned int n_samples_, unsigned int n_classes_, unsigned int n_features_) {

    unsigned int feat_col = threadIdx.x + (blockIdx.x * blockDim.x);

    // End condition check
    if (feat_col < n_features_) { 
        // Unroll loop for efficiency
        for (unsigned int i = 0; i < n_classes_; ++i) {
            feature_probs[RM_Index(i, feat_col, n_features_)] /= class_count_[i];
        
            // Perform this division only once per class
            if (feat_col == 0) {
                class_count_[i] /= (float)n_samples_;
            }
        }
    }
}