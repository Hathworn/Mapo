#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
    return (row * width + col);
}

__global__ void BernoulliNBCalcKernel(const float *d_data, const int *d_labels, float *feature_probs, float *class_count_, unsigned int n_samples_, unsigned int n_classes_, unsigned int n_features_) {

    unsigned int feat_col = blockIdx.x * blockDim.x + threadIdx.x;

    if (feat_col < n_features_) { // Check if thread should execute
        unsigned int row;

        for (unsigned int i = 0; i < n_samples_; ++i) {
            row = d_labels[i];

            // Accumulate feature probabilities
            atomicAdd(&feature_probs[RM_Index(row, feat_col, n_features_)],
                      d_data[RM_Index(i, feat_col, n_features_)]);

            if (feat_col == 0) {
                // Safeguard class count with atomic
                atomicAdd(&class_count_[row], 1);
            }
        }
    }
}