#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
    return (row * width + col);
}

__global__ void MultinomialNBCalcKernel(const float *d_data, const int *d_labels, float *feature_probs, float *class_priors, unsigned int n_samples_, unsigned int n_classes_, unsigned int n_features_) {

    // Calculate unique thread index
    unsigned int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure the current thread deals only with available features
    if (tidx < n_features_) {
        for (unsigned int i = 0; i < n_samples_; ++i) {
            unsigned int row = d_labels[i];
            
            // Each thread independently updates its feature probability index
            atomicAdd(&feature_probs[RM_Index(row, tidx, n_features_)], d_data[RM_Index(i, tidx, n_features_)]);

            // Thread 0 moderates class prior updating
            if (tidx == 0) {
                atomicAdd(&class_priors[row], 1);
            }
        }
    }
}