#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
    return (row * width + col);
}

__global__ void GaussianNBMeanKernel(float *feature_means_, int *class_count_, float *class_priors_, unsigned int n_samples_, unsigned int n_classes_, unsigned int n_features_) {
    unsigned int tidx = threadIdx.x;
    unsigned int feat_col = tidx + (blockIdx.x * blockDim.x);

    if (feat_col < n_features_) {
        // Cache class counts in shared memory to reduce global memory accesses
        __shared__ int shared_class_count[MAX_CLASSES]; // Assuming MAX_CLASSES is defined suitably
        if (tidx < n_classes_) {
            shared_class_count[tidx] = class_count_[tidx];
        }
        __syncthreads();

        // Calculate Means
        for (unsigned int i = 0; i < n_classes_; ++i) {
            feature_means_[RM_Index(i, feat_col, n_features_)] /= shared_class_count[i];
        }

        // Calculate Class Priors
        if (feat_col == 0) {
            for (unsigned int i = 0; i < n_classes_; ++i) {
                class_priors_[i] = (float)shared_class_count[i] / n_samples_;
            }
        }
    }
}