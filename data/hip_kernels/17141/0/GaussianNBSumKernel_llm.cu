#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
    return (row * width + col);
}

__global__ void GaussianNBSumKernel(const float *d_data, const int *d_labels, float *feature_means_, int *class_count_, unsigned int n_samples_, unsigned int n_classes_, unsigned int n_features_) {
    // Optimize by using shared memory to reduce global memory accesses
    extern __shared__ int shared_class_count[];

    unsigned int tidx = threadIdx.x;
    unsigned int feat_col = tidx + (blockIdx.x * blockDim.x);
    unsigned int i = 0, row = 0;

    // Initialize shared memory for class counts
    if (tidx < n_classes_ && feat_col < n_features_) {
        shared_class_count[tidx] = 0;
    }
    __syncthreads();

    if (feat_col < n_features_) { // End condition check
        float mean_sum = 0.0f; // Local accumulator for feature means

        for (i = 0; i < n_samples_; ++i) { // For each training sample
            row = d_labels[i];

            // Accumulate feature mean locally
            mean_sum += d_data[RM_Index(i, feat_col, n_features_)];

            // Accumulate class count in shared memory
            if (feat_col == 0) {
                atomicAdd(&shared_class_count[row], 1);
            }
        }

        // Store back calculated mean sum to global memory
        feature_means_[RM_Index(row, feat_col, n_features_)] = mean_sum;

        __syncthreads();

        // Store class counts from shared to global memory by a single thread per block
        if (tidx == 0) {
            for (int c = 0; c < n_classes_; ++c) {
                atomicAdd(&class_count_[c], shared_class_count[c]);
            }
        }
    }
    return;
}