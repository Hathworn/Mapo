#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
    return (row * width + col);
}

__global__ void ComplementNBNormalizeKernel(float *feature_weights_, float *per_class_sum_, unsigned int n_classes_, unsigned int n_features_) {
    // Calculate the global feature index
    int feat_col = blockIdx.x * blockDim.x + threadIdx.x;

    // Use a shared memory to cache per_class_sum_ for faster access if n_classes_ size is relatively small
    extern __shared__ float shared_class_sum[];

    if (threadIdx.x < n_classes_) {
        shared_class_sum[threadIdx.x] = per_class_sum_[threadIdx.x];
    }
    __syncthreads(); // Ensure all shared memory is loaded

    if (feat_col < n_features_) { // Boundary condition check
        for (unsigned int i = 0; i < n_classes_; ++i) { // For each class
            // Perform division
            feature_weights_[RM_Index(i, feat_col, n_features_)] /= shared_class_sum[i];
        }
    }
}