#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
    return (row * width + col);
}

__global__ void ComplementNBCalcKernel(const float *d_data, const int *d_labels, float *per_class_feature_sum_, float *per_feature_sum_, unsigned int n_samples_, unsigned int n_features_) {

    // Calculate global thread ID 
    unsigned int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    
    if (tidx < n_features_) {  // Check if within feature bounds
        float class_sum = 0.0f;  // Local cache for class sum
        float feature_sum = 0.0f;  // Local cache for feature sum

        // For each document/sample
        for (unsigned int i = 0; i < n_samples_; ++i) {
            unsigned int row = d_labels[i];
            float value = d_data[RM_Index(i, tidx, n_features_)];

            // Accumulate in local variables
            class_sum += value;
            feature_sum += value;
        }

        // Write back results
        per_class_feature_sum_[RM_Index(row, tidx, n_features_)] = class_sum;
        per_feature_sum_[tidx] = feature_sum;
    }
}
```
