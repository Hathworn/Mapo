#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ inline unsigned int RM_Index(unsigned int row, unsigned int col, unsigned int width) {
    return (row * width + col);
}

__global__ void ComplementNBLearnKernel(float *feature_weights_, float *per_class_feature_sum_, float *per_feature_sum_, float *per_class_sum_, float all_sum_, unsigned int n_classes_, unsigned int n_features_) {
    // Use shared memory to reduce global memory access
    extern __shared__ float shared_mem[];
    
    unsigned int tidx = threadIdx.x;
    int feat_col = tidx + (blockIdx.x * blockDim.x);

    if (feat_col < n_features_) { // Boundary check
        float *s_per_feature_sum = shared_mem;
        
        // Load per_feature_sum_ into shared memory
        if (tidx < n_features_) {
            s_per_feature_sum[tidx] = per_feature_sum_[tidx];
        }
        __syncthreads(); // Synchronize to ensure shared memory is populated

        float den_sum = 0;
        float num_sum = 0;

        for (unsigned int i = 0; i < n_classes_; ++i) { // For each class
            den_sum = all_sum_ - per_class_sum_[i];
            num_sum = s_per_feature_sum[feat_col] - per_class_feature_sum_[RM_Index(i, feat_col, n_features_)];

            feature_weights_[RM_Index(i, feat_col, n_features_)] =
                log(num_sum + 1.0) - log(den_sum + n_features_);
        }
    }
}