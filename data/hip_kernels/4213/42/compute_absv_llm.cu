#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void compute_absv(const unsigned int nSpheres, const float* velX, const float* velY, const float* velZ, float* d_absv) {
    unsigned int my_sphere = blockIdx.x * blockDim.x + threadIdx.x;
    if (my_sphere < nSpheres) {
        // Direct computation without extra array 'v'
        float vx = velX[my_sphere];
        float vy = velY[my_sphere];
        float vz = velZ[my_sphere];
        d_absv[my_sphere] = sqrtf(vx * vx + vy * vy + vz * vz); // Use sqrtf for float optimization
    }
}