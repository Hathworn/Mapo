#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kAdd_contiguous(float* a, float* b, float* dest, unsigned int numEls) {
    // Calculate global thread index
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Loop unrolling for improved memory access
    if (idx < numEls) {
        dest[idx] = a[idx] + b[idx];
    }
}