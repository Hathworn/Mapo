#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int position;  // index of the largest value
__device__ int largest;   // value of the largest value
int lenString = 593;
int maxNumStrings = 1000000;
int threshold = 2;

__global__ void compare(char *d_a, int *d_b, int *d_c, int size, int lenString, int threshold) {
    int my_id = blockDim.x * blockIdx.x + threadIdx.x;

    if (my_id < size) {
        int diffs = 0;

        if (my_id == position) {
            d_c[my_id] = 2;
        } else if (d_c[my_id] == 0) {
            // Optimize loop access by working with pointers directly
            char *target = d_a + (lenString * position);
            char *current = d_a + (my_id * lenString);

            for (int x = 0; x < lenString; x++) {
                diffs += (bool)(target[x] ^ current[x]);
                if (diffs > threshold) 
                    break;  // Exit early if threshold is exceeded
            }

            if (diffs <= threshold) {
                atomicAdd(&d_b[position], d_b[my_id]);  // Ensure atomic operation for thread safety
                d_c[my_id] = 1;
            }
        }
    }
}