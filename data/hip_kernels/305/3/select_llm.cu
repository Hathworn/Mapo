#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int position;            // index of the largest value
__device__ int largest;             // value of the largest value
int lenString = 593;
int maxNumStrings = 1000000;
int threshold = 2;

__device__ void cuda_select(int *db, int size) {
    int my_id = blockDim.x * blockIdx.x + threadIdx.x;

    if(my_id < size) {
        int idx1 = 2 * my_id;
        int idx2 = idx1 + 1;
        db[my_id] = db[idx1] > db[idx2] ? db[idx1] : db[idx2]; // Use conditional operator
    }
}

__global__ void select(int *db, int size) {
    extern __shared__ int shared_db[]; // Use shared memory for increased performance
    int tid = threadIdx.x;
    int numThreads = blockDim.x;
    int height = (int)ceil(log2((double)size));
    
    for(int i = 0; i < height; i++) {
        if(tid < size) {
            shared_db[tid] = db[tid];
        }
        __syncthreads();  // Synchronize threads
        
        if(tid < size / 2) {
            int idx1 = 2 * tid;
            int idx2 = idx1 + 1;
            shared_db[tid] = shared_db[idx1] > shared_db[idx2] ? shared_db[idx1] : shared_db[idx2];
        }
        __syncthreads();

        if(tid < size / 2) {
            db[tid] = shared_db[tid];
        }
        size = (int)ceil((double)size / 2);
    }
    
    if(tid == 0) {
        largest = db[0];
    }
}