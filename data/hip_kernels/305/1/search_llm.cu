#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int position;			// index of the largest value
__device__ int largest;				// value of the largest value
int lenString = 593;
int maxNumStrings = 1000000;
int threshold = 2;

__global__ void search(int *d_b, int *d_c, int size) {
    int my_id = blockDim.x * blockIdx.x + threadIdx.x;
    if (my_id < size) {  // Ensure my_id is within bounds
        if ((d_c[my_id] == 0) && (d_b[my_id] == largest)) {
            position = my_id;
        }
    }
}