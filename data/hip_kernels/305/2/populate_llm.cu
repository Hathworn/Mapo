#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int position;            // index of the largest value
__device__ int largest;             // value of the largest value
int lenString = 593;
int maxNumStrings = 1000000;
int threshold = 2;

__global__ void populate(int *d_b, int *copy_db, int *d_c, int size, int *left) {

    // Use shared memory for efficiency if needed (not shown here as it's not necessary for current ops)
    extern __shared__ int shared_mem[];

    // Calculate unique thread index
    int my_id = blockDim.x * blockIdx.x + threadIdx.x;

    // Initialize left only once by the first thread
    if (my_id == 0) *left = 1;    

    // Ensure only valid threads access memory
    if (my_id < size) {
        // Perform operations and update the copy
        int n = abs((bool)d_c[my_id] - 1);
        copy_db[my_id] = d_b[my_id] * n;
    }
}