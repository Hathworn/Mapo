#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int position;            // index of the largest value
__device__ int largest;             // value of the largest value
int lenString = 593;
int maxNumStrings = 1000000;
int threshold = 2;

__global__ void anyLeft(int *d_c, int *remaining, int size) {
    // Calculate unique thread index
    int my_id = blockDim.x * blockIdx.x + threadIdx.x;

    // Check bounds and update only if it remains greater
    if (my_id < size && d_c[my_id] == 0) {
        // Atomically set remaining to 0 to ensure only one operation
        atomicExch(remaining, 0);
    }
}