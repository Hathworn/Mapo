#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized global kernel function to use shared memory and reduce loop overhead
__global__ void NormalizeOutput(const int num_elements, const int* __restrict__ original, int64_t* __restrict__ to_normalize, int64_t batch_index, int64_t class_index) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_elements) {
        int index3 = idx * 3;
        to_normalize[index3] = batch_index;
        to_normalize[index3 + 1] = class_index;
        to_normalize[index3 + 2] = static_cast<int64_t>(original[idx]);
    }
}