#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void setup_state(hiprandState* state, unsigned long long seed) {
  // Use built-in variables for thread indexing for potential parallel state setup
  int idx = blockIdx.x * blockDim.x + threadIdx.x; 
  hiprand_init(seed, idx, 0, &state[idx]);
}