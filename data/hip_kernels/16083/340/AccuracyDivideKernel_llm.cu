#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use thread id 0 for computation to reduce atomic contention
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *accuracy /= N;
    }
}