#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void square_matrix_kernel(int32_t num_rows, int32_t num_cols, const float* feats, int32_t ldf, float* feats_sq, int32_t lds) {
    // Use shared memory to improve access speed
    extern __shared__ float shared_feats[];
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < num_rows && col < num_cols) {
        // Load element into shared memory
        shared_feats[threadIdx.y * blockDim.x + threadIdx.x] = feats[row * ldf + col];
        __syncthreads();
        
        // Calculate square using shared memory
        feats_sq[row * lds + col] = shared_feats[threadIdx.y * blockDim.x + threadIdx.x] * shared_feats[threadIdx.y * blockDim.x + threadIdx.x];
    }
}