#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x + blockIdx.x * blockDim.x; // Calculate global thread index
    int half_length = row_length / 2;

    while (thread_id < half_length) { // Loop across the row using global thread index
        if (thread_id != 0) { // Ignore special case for index 0
            float2 val = reinterpret_cast<float2 *>(A_in + blockIdx.x * ldi)[thread_id];
            float ret = val.x * val.x + val.y * val.y;

            A_out[blockIdx.x * ldo + thread_id] = use_power ? ret : sqrtf(ret);
        }
        thread_id += gridDim.x * blockDim.x; // Move to the next element
    }

    __shared__ bool is_first_thread_done;
    if (threadIdx.x == 0) { // Ensures only one thread handles the special boundary case
        float *Ar = A_in + blockIdx.x * ldi;
        float *Aw = A_out + blockIdx.x * ldo;
        float real = Ar[0];
        float im = Ar[row_length];

        Aw[0] = use_power ? real * real : fabs(real);
        Aw[half_length] = use_power ? im * im : fabs(im);
        
        is_first_thread_done = true;
    }
}