#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void power_spectrum_kernel(int row_length, const float *A_in, int32_t ldi, float *A_out, int32_t ldo, bool use_power) {
    int thread_id = threadIdx.x;
    int block_id = blockIdx.x;
    const float *Ar = A_in + block_id * ldi;
    float *Aw = A_out + block_id * ldo;

    int half_length = row_length >> 1; // Use bit shift for division by 2

    // Use simplified indexing and loop unrolling for better performance
    for (int idx = thread_id; idx < half_length; idx += blockDim.x) {
        float2 val = reinterpret_cast<const float2 *>(Ar)[idx];
        float ret = val.x * val.x + val.y * val.y;
        Aw[idx] = use_power ? ret : sqrtf(ret);
    }

    __syncthreads(); // Ensure all threads have completed processing

    // Handle special cases with all threads to avoid branch divergence
    if (thread_id < 2) {
        float real = Ar[0];
        float im = Ar[row_length];
        if (thread_id == 0) {
            Aw[0] = use_power ? (real * real) : fabs(real);
        } else if (thread_id == 1) {
            Aw[half_length] = use_power ? (im * im) : fabs(im);
        }
    }
}