#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use atomic operation for safe concurrent access to shared memory
    atomicExch(accuracy, *accuracy / N);
}