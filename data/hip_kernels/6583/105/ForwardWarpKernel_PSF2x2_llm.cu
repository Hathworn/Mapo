#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Compute target pixel's float coordinates
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    // Split target coordinates into integer and fractional parts
    float px, py, dx, dy;
    dx = modff(cx, &px);
    dy = modff(cy, &py);

    // Integer target pixel coordinates
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);
    float value = src[image_row_offset + j];
    
    // Atomic update only if target coordinates are within bounds
    float weight;
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = dx * dy;
        atomicAdd(&dst[ty * image_stride + tx], value * weight);
        atomicAdd(&normalization_factor[ty * image_stride + tx], weight);
    }
    
    tx -= 1; // Bottom-left corner
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = (1.0f - dx) * dy;
        atomicAdd(&dst[ty * image_stride + tx], value * weight);
        atomicAdd(&normalization_factor[ty * image_stride + tx], weight);
    }

    ty -= 1; // Upper-left corner
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(&dst[ty * image_stride + tx], value * weight);
        atomicAdd(&normalization_factor[ty * image_stride + tx], weight);
    }

    tx += 1; // Upper-right corner
    if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
        weight = dx * (1.0f - dy);
        atomicAdd(&dst[ty * image_stride + tx], value * weight);
        atomicAdd(&normalization_factor[ty * image_stride + tx], weight);
    }
}