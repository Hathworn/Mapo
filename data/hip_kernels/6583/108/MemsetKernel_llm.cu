#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}
__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    // Calculate the global thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Flatten the 2D index to 1D
    int pos = idy * w + idx;

    // Boundary check
    if (idy < h && idx < w)
    {
        // Set the image value at position
        image[pos] = value;
    }
}