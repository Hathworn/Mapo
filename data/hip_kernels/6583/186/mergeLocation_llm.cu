#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate the index using block and thread identifiers
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Use shared memory for improved memory access efficiency
        __shared__ short2 shared_loc[256];  // Adjust size as needed

        // Load location into shared memory
        shared_loc[threadIdx.x] = loc_[ptidx];
        __syncthreads();

        // Read from shared memory
        x[ptidx] = shared_loc[threadIdx.x].x * scale;
        y[ptidx] = shared_loc[threadIdx.x].y * scale;
    }
}