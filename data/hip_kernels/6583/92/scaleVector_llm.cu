#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void scaleVector(float *d_res, const float *d_src, float scale, const int len)
{
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;

    // Use branchless conditional within valid range to avoid warp divergence
    if (pos < len) {
        d_res[pos] = d_src[pos] * scale;
    }
}