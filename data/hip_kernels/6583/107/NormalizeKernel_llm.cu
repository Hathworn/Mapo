#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    // Calculate two-dimensional grid stride.
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Index to access array elements in a flattened manner.
    int stride_j = blockDim.x * gridDim.x;
    int stride_i = blockDim.y * gridDim.y;

    // Stride loop to handle larger input data sets.
    for (int x = i; x < h; x += stride_i) {
        for (int y = j; y < w; y += stride_j) {

            const int pos = x * s + y;

            float scale = normalization_factor[pos];
            float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

            image[pos] *= invScale;
        }
    }
}