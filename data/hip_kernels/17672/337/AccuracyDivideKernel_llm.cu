#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Each thread computes part of the division
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        float threadAcc = accuracy[idx];
        accuracy[idx] = threadAcc / N;
    }
}