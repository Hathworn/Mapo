#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"


using namespace std;

#define MAX_N_TERMS 10

__global__ void MC_Integratev2(float* degrees_g, int dimension, int n_terms, float* I_val, long int seed, int thread_max_iterations) {
    // Get the global and local ids
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int lid = threadIdx.x;

    float I = 0.0f;
    float f[MAX_N_TERMS];

    // Dynamically allocate shared memory for 'degrees' and 'I_shared'
    extern __shared__ float shared_mem[];
    float* I_shared = shared_mem;
    I_shared[0] = 0;
    float* degrees = &shared_mem[1];

    // Initialize the local copy of 'degrees' for the shared copy
    if (lid < (2 + n_terms) * dimension) {
        // Copy one element of degrees
        degrees[lid] = degrees_g[lid];
    }

    // Create a state in private memory
    hiprandState state;
    // Initialize the random number generator
    hiprand_init(seed, id, 0, &state);

    // Synchronize all threads to assure that 'degrees' is initialized
    __syncthreads();

    for (int iter_count = 0; iter_count < thread_max_iterations; iter_count++) {
        // Initialize f with the coefficients
        for (int term_i = 0; term_i < n_terms; term_i++) {
            f[term_i] = degrees[(2 + term_i) * dimension];
        }

        for (int d = 1; d < dimension; d++) {
            // Generate a random number in the range of the limits of this dimension
            float x = hiprand_uniform(&state);  // x between 0 and 1
            // Generate dimension sample based on the limits of the dimension
            x = x * (degrees[1 * dimension + d] - degrees[0 * dimension + d]) + degrees[0 * dimension + d];
            for (int term_i = 0; term_i < n_terms; term_i++) {
                // Multiply f of this term by x^(power of this dimension in this term)
                f[term_i] *= powf(x, degrees[(2 + term_i) * dimension + d]); // Use powf for faster computation
            }
        }

        // Add the evaluation to the private summation
        for (int term_i = 0; term_i < n_terms; term_i++) {
            I += f[term_i];
        }
    }
    
    // Use atomicAdd for shared memory and global memory reductions
    atomicAdd(I_shared, I);
    __syncthreads(); // Synchronize threads

    if (lid == 0) {
        atomicAdd(I_val, *I_shared);
    }
}