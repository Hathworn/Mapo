#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i_start = threadIdx.y + blockDim.y * blockIdx.y;
    int stride = blockDim.y * gridDim.y;

    // Optimize by processing multiple rows per thread
    for (int i = i_start; i < h; i += stride) {
        if (j < w) {
            const int pos = i * w + j;
            image[pos] = value;
        }
    }
}