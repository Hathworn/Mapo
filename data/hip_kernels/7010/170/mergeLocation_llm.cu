#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Use a shared memory buffer to reduce memory latency
    extern __shared__ short2 shared_loc[];
    
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Load data into shared memory
        shared_loc[threadIdx.x] = loc_[ptidx];
        __syncthreads();

        // Perform computations using shared memory
        short2 loc = shared_loc[threadIdx.x];
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}