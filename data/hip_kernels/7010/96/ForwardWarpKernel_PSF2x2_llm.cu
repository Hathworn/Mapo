#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // bottom left corner of a target pixel
    float cx = u[flow_row_offset + j] * time_scale + static_cast<float>(j) + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + static_cast<float>(i) + 1.0f;

    // pixel containing bottom left corner
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    // target pixel integer coords
    int tx = static_cast<int>(px);
    int ty = static_cast<int>(py);
    
    float value = src[image_row_offset + j];
    float weight;

    // Combine multiple boundary checks using a single var to reduce redundant checks
    bool pixelInBounds;

    // fill pixel containing bottom right corner
    pixelInBounds = !(tx >= w || tx < 0 || ty >= h || ty < 0);
    if (pixelInBounds)
    {
        weight = dx * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // fill pixel containing bottom left corner
    tx -= 1;
    pixelInBounds = !(tx >= w || tx < 0 || ty >= h || ty < 0);
    if (pixelInBounds)
    {
        weight = (1.0f - dx) * dy;
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // fill pixel containing upper left corner
    ty -= 1;
    pixelInBounds = !(tx >= w || tx < 0 || ty >= h || ty < 0);
    if (pixelInBounds)
    {
        weight = (1.0f - dx) * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }

    // fill pixel containing upper right corner
    tx += 1;
    pixelInBounds = !(tx >= w || tx < 0 || ty >= h || ty < 0);
    if (pixelInBounds)
    {
        weight = dx * (1.0f - dy);
        atomicAdd(dst + ty * image_stride + tx, value * weight);
        atomicAdd(normalization_factor + ty * image_stride + tx, weight);
    }
}