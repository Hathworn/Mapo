#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void KCInitRNGStates(const uint32_t* gSeeds, hiprandStateMRG32k3a_t* gStates, size_t totalCount)
{
for(uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;
threadId < totalCount;
threadId += (blockDim.x * gridDim.x))
{
hiprand_init(gSeeds[threadId], threadId, 0, &gStates[threadId]);
}
}