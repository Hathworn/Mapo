#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void KCInitRNGStates(const uint32_t* gSeeds, hiprandStateMRG32k3a_t* gStates, size_t totalCount)
{
    // Calculate global thread index
    uint32_t threadId = threadIdx.x + blockDim.x * blockIdx.x;

    // Ensure threadId is within totalCount
    if(threadId < totalCount) {
        // Initialize RNG state for the current thread
        hiprand_init(gSeeds[threadId], threadId, 0, &gStates[threadId]);
    }
}
```
