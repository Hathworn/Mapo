#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    // Calculate thread id using efficient bitwise operations
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    
    // Use a loop to handle more data per thread if necessary
    for (int i = tid; i < N; i += gridDim.x * blockDim.x) {
        out[i] = in[i] > 0 ? in[i] : 0;
    }
}