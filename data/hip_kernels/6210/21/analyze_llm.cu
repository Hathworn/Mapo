#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void analyze(const float *input, float *sum, int numElements) {
    // Use shared memory to minimize global memory accesses
    extern __shared__ float sharedSum[];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = threadIdx.x;

    sharedSum[tid] = (i < numElements) ? input[i] : 0.0f;
    __syncthreads();

    // Perform parallel reduction in shared memory
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sharedSum[tid] += sharedSum[tid + s];
        }
        __syncthreads();
    }

    // Use atomic operation to reduce partial sums from each block
    if (tid == 0) {
        atomicAdd(sum, sharedSum[0]);
    }
}