#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void init(uint32_t *v, uint32_t val, uint32_t n) {
    // Use register for faster access
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;

    // Unrolling the loop
    uint32_t gridStride = blockDim.x * gridDim.x; 
    for (uint32_t idx = i; idx < n; idx += gridStride) {
        v[idx] = val;
    }

    // Print only once per grid, using blockIdx and threadIdx
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        printf("init\n");
    }
}