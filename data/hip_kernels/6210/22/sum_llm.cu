#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sum(const float *input, float *output, int numElements) {
    // Use shared memory for intermediate sums
    extern __shared__ float sharedData[];
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;
    
    sharedData[tid] = (idx < numElements) ? input[idx] : 0.f;
    __syncthreads();

    // Reduce sum in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            sharedData[tid] += sharedData[tid + stride];
        }
        __syncthreads();
    }

    // Write result of this block's reduction to output
    if (tid == 0) {
        atomicAdd(output, sharedData[0]);
    }
}