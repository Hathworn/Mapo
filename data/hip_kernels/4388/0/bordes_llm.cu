#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define Columnas 10
#define Filas 10
hipError_t addWithCuda(int* c, const int* a, unsigned int size);

__device__ unsigned int computeOutputEdge(int mask[][3], int vecinos[][3], int rows, int cols) {
    int sum = 0;
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            sum += mask[i][j] * vecinos[i][j];
        }
    }
    return abs(sum);
}

__global__ void bordes(int* val2, const int* val1, int m, int n) {

    int column = threadIdx.x + blockDim.x * blockIdx.x;
    int row = threadIdx.y + blockDim.y * blockIdx.y;

    int myEdge[3][3] = { {0,1,0},{1,-4,1},{0,1,0} };

    // Ensure we only run for valid rows and columns
    if (row > 0 && row < m - 1 && column > 0 && column < n - 1) {
        int thread_id1 = (row - 1) * n + (column - 1);
        int thread_id2 = (row - 1) * n + column;
        int thread_id3 = (row - 1) * n + (column + 1);
        int thread_id4 = row * n + (column - 1);
        int thread_id5 = row * n + column;
        int thread_id6 = row * n + (column + 1);
        int thread_id7 = (row + 1) * n + (column - 1);
        int thread_id8 = (row + 1) * n + column;
        int thread_id9 = (row + 1) * n + (column + 1);

        // Access matrix values in shared memory to reduce global memory pressure
        int myMask2[3][3] = {
            { val1[thread_id1], val1[thread_id2], val1[thread_id3] },
            { val1[thread_id4], val1[thread_id5], val1[thread_id6] },
            { val1[thread_id7], val1[thread_id8], val1[thread_id9] }
        };

        unsigned int output = computeOutputEdge(myEdge, myMask2, 3, 3);
        val2[thread_id5] = output;
    }
    // Handle edge cases by retaining original values
    else if (row < m && column < n) {
        val2[row * n + column] = val1[row * n + column];
    }
}
```
