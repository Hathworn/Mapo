#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sub3(float *val1, float *val2, int *num_elem)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global thread index
    if (i < *num_elem) { // Ensure thread does not exceed vector bounds
        val1[i] += val2[i] + 1;
    }
}