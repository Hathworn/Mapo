#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ unsigned int doIterations(double const realPart0, double const imagPart0, unsigned int const maxIters) {
    // Initialise: z = z0
    double realPart = realPart0;
    double imagPart = imagPart0;
    unsigned int count = 0;
    // Loop until escape
    while ((count <= maxIters) && (realPart*realPart + imagPart*imagPart <= 4.0)) {
        ++count;
        // Update: z = z*z + z0;
        double const oldRealPart = realPart;
        realPart = realPart * realPart - imagPart * imagPart + realPart0;
        imagPart = 2.0 * oldRealPart * imagPart + imagPart0;
    }
    return count;
}

__device__ size_t calculateGlobalIndex() {
    // Calculate global thread index directly
    size_t const globalThreadIdx = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (blockIdx.x + blockIdx.y * gridDim.x));
    return globalThreadIdx;
}

__global__ void processMandelbrotElement(double *out, const double *x, const double *y, const unsigned int maxIters, const unsigned int numel) {
    // Calculate global thread index
    size_t const globalThreadIdx = calculateGlobalIndex();

    // Check thread bounds
    if (globalThreadIdx >= numel) return;

    // Process coordinates and execute iterations
    double const realPart0 = x[globalThreadIdx];
    double const imagPart0 = y[globalThreadIdx];
    unsigned int const count = doIterations(realPart0, imagPart0, maxIters);
    out[globalThreadIdx] = log(double(count + 1));
}