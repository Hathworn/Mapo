#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void add3(float *val1, float *val2, int *num_elem)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Calculate the global index
    int total_threads = gridDim.x * blockDim.x; // Calculate total threads

    for (int idx = i; idx < *num_elem; idx += total_threads) { // Loop to cover all elements
        val1[idx] += val2[idx]; // Perform addition
    }
}