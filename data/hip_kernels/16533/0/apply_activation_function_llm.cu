#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ float activation_function(float x)
{
    return 1 / (1 + exp(-x));
}

__global__ void apply_activation_function(float *input, float *output, const int N)
{
    // Calculate global thread index
    const int idx = blockIdx.x * blockDim.x + threadIdx.x; 
    
    // Process only valid indices
    if (idx < N) {
        output[idx] = activation_function(input[idx]);
    }
}