#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void makeError(float *err, float *output, unsigned int Y, const int N)
{
    // Calculate global thread ID
    const int globalId = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate stride for the grid
    const int stride = blockDim.x * gridDim.x;

    // Loop with stride
    for (int idx = globalId; idx < N; idx += stride) {
        // Compute error with conditional assignment
        err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]);
    }
}