#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void apply_grad(float *output, float *grad, const int N)
{
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int stride = blockDim.x * gridDim.x;

    // Use a stride loop to improve performance and maximize resource usage
    for (int i = idx; i < N; i += stride) {
        output[i] += dt * grad[i];
    }
}