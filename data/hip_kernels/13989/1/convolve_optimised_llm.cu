#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

const int ITERS = 500;

__global__ void convolve_optimised(float* data_in, float* data_out, float* kernel, int kernelSize, int BLOCK_SIZE)
{
    int tx = threadIdx.x;
    int bk = blockIdx.x;

    extern __shared__ float data_in_shared[];

    int pos = (bk * BLOCK_SIZE) + tx;

    // Load input data into shared memory, handle boundary condition for shared memory
    data_in_shared[tx] = data_in[pos];
    if (tx < kernelSize - 1) {
        data_in_shared[BLOCK_SIZE + tx] = data_in[min(pos + BLOCK_SIZE, (blockDim.x * gridDim.x) - 1)];
    }
    __syncthreads();

    // Initialize output value
    float sum = 0.0f;

    // Perform convolution
    for (int i = 0; i < kernelSize; i++) {
        sum += kernel[i] * data_in_shared[tx + i];
    }

    // Write result to global memory
    data_out[pos] = sum;
}