#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

const int ITERS = 500;

__global__ void convolve(float* data_in, float* data_out, float* kernel, int kernelSize, int BLOCK_SIZE)
{
    int tx = threadIdx.x;
    int bk = blockIdx.x;
    int pos = (bk * BLOCK_SIZE) + tx;

    // Use shared memory for kernel to reduce global memory access
    extern __shared__ float shared_kernel[];

    if (tx < kernelSize) {
        shared_kernel[tx] = kernel[tx];
    }
    __syncthreads();

    data_out[pos] = 0;

    for (int i = 0; i < kernelSize; i++) {
        if (pos - i >= 0) {
            data_out[pos] += shared_kernel[i] * data_in[pos - i];  // Use shared memory
        }
    }
}