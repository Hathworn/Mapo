#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void scan(int *v, const int n)
{
    int tIdx = threadIdx.x;
    int step = 1;

    // Optimize: Use shared memory for faster access
    __shared__ int temp[1024];

    if (tIdx < n) {
        temp[tIdx] = v[tIdx];
    }
    __syncthreads();

    while (step < n) {
        int indiceDroite = tIdx;
        int indiceGauche = indiceDroite + step;

        if (indiceGauche < n) {
            // Optimize: Accumulate values in shared memory
            temp[indiceDroite] += temp[indiceGauche];
        }

        step *= 2;
        __syncthreads();
    }

    if (tIdx < n) {
        // Write back to global memory
        v[tIdx] = temp[tIdx];
    }
}