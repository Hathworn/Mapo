#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void makeFlist(unsigned int *d_trans_offset, unsigned int *d_transactions, unsigned int *d_flist, unsigned int num_transactions, unsigned int num_items_in_transactions) {

    __shared__ unsigned int private_items[max_unique_items];

    int tx = threadIdx.x;
    int index = tx + blockDim.x * blockIdx.x;
    int location_x;

    // Initialize shared memory efficiently
    for (int i = tx; i < max_unique_items; i += blockDim.x) {
        private_items[i] = 0;
    }

    __syncthreads();

    // Efficient atomic addition only if index is valid
    if (index < num_items_in_transactions && d_transactions[index] < max_unique_items) {
        atomicAdd(&private_items[d_transactions[index]], 1);
    }

    __syncthreads();

    // Efficient accumulation into global memory
    for (int i = tx; i < max_unique_items; i += blockDim.x) {
        atomicAdd(&d_flist[i], private_items[i]);
    }
}