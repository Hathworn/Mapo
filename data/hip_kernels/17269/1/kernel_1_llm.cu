#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_1(int columns, int rows, float* mat1, float* matanswer) {
    int columna = threadIdx.x;  // Thread index represents the column
    float temp_value = 0;
    
    // Unrolling loop for better performance
    int k = 0;
    for (; k + 3 < rows; k += 4) {
        temp_value += mat1[(k * columns) + columna] +
                      mat1[((k + 1) * columns) + columna] +
                      mat1[((k + 2) * columns) + columna] +
                      mat1[((k + 3) * columns) + columna];
    }

    // Handle remaining iterations
    for (; k < rows; k++) {
        temp_value += mat1[(k * columns) + columna];
    }
    
    matanswer[columna] = temp_value;
}