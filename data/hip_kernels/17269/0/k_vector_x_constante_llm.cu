#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

using namespace std;

void print_function(int*, int);

// Programando Tarjeta//

__global__ void k_vector_x_constante(int* arr, int* answer, int n, int k) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Use shared memory for caching if array size permits
    extern __shared__ int shared_mem[];
    if (idx < n) {
        shared_mem[threadIdx.x] = arr[idx];  // Load into shared memory
        __syncthreads();  // Synchronize to ensure all threads have loaded their elements
        answer[idx] = shared_mem[threadIdx.x] * k;  // Perform computation
    }
}