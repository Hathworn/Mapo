#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_1(int columns, int rows, float* mat1, float* matanswer) {
    int columna = threadIdx.x;  // Column index
    if (columna < columns) { // Bounds check for safety
        float temp_value = 0;
        for (int k = 0; k < rows; ++k) {
            temp_value += mat1[(k * columns) + columna];
        }
        matanswer[columna] = temp_value;
    }
}