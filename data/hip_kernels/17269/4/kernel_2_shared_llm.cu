#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_2_shared(int columns, int rows, float* mat1, float* matanswer) {
    __shared__ float temp_answer[32];

    // Initialize shared memory more efficiently
    if (threadIdx.y == 0 && threadIdx.x < 32) {
        temp_answer[threadIdx.x] = 0.0f;
    }
    __syncthreads();  // Ensure all initializations are complete

    int thread_mat_colid = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_mat_rowid = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (thread_mat_colid < columns && thread_mat_rowid < rows) {
        int position_in_matrix = thread_mat_rowid * columns + thread_mat_colid;
        
        // Use block wide reduction in shared memory
        atomicAdd(&(temp_answer[threadIdx.x]), mat1[position_in_matrix]);
        
        if (blockIdx.x == 0 && blockIdx.y == 1 && threadIdx.x == 0 && threadIdx.y == 0) {
            printf("%d Value %f \n", position_in_matrix, mat1[position_in_matrix]);
        }
    }
    __syncthreads();  // Ensure all atomic additions are complete

    // Write results from shared memory to global memory only when necessary
    if (threadIdx.y == 0 && thread_mat_colid < columns) {
        atomicAdd(&(matanswer[thread_mat_colid]), temp_answer[threadIdx.x]);
    }
}