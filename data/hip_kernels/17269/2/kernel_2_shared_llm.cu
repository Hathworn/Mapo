#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kernel_2_shared(int columns, int rows, float* mat1, float* matanswer) {
    // Use shared memory for temporary storage
    __shared__ float temp_answer[32];
    
    // Initialize shared memory only by the first row of threads
    if (threadIdx.y == 0) {
        temp_answer[threadIdx.x] = 0.0f;
    }
    __syncthreads();

    int thread_mat_colid = blockIdx.x * blockDim.x + threadIdx.x;
    int thread_mat_rowid = blockIdx.y * blockDim.y + threadIdx.y;
    int position_in_matrix = thread_mat_rowid * columns + thread_mat_colid;

    // Each thread accumulates its matrix element into shared memory
    if (thread_mat_colid < columns && thread_mat_rowid < rows) {
        atomicAdd(&(temp_answer[threadIdx.x]), mat1[position_in_matrix]);
    }
    __syncthreads();

    // Check specific condition for debug print
    if (blockIdx.x == 1 && blockIdx.y == 0 && threadIdx.x == 0 && threadIdx.y == 0) {
        printf("%d Value %f \n", position_in_matrix, mat1[position_in_matrix]);
    }

    // Update the global matrix answer only by the last row of threads
    if (threadIdx.y == blockDim.y - 1) {
        atomicAdd(&(matanswer[thread_mat_colid]), temp_answer[threadIdx.x]);
    }
}