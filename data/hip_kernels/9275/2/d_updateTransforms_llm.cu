#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void d_updateTransforms(float* d_currentTransform, float3* d_cameraPosition)
{
    // Efficiently update camera position using shared memory
    __shared__ float transform[12];
    if (threadIdx.x < 12) 
    {
        transform[threadIdx.x] = d_currentTransform[threadIdx.x];
    }
    __syncthreads(); // Synchronize threads to ensure shared memory is populated

    if (threadIdx.x == 0) 
    {
        d_cameraPosition->x = transform[3];
        d_cameraPosition->y = transform[7];
        d_cameraPosition->z = transform[11];
    }
}