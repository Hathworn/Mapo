#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void d_addToCurrentTransform(float* d_currentTransform, float* d_invViewMatrix) {
    // Use shared memory for faster access
    __shared__ float invViewMatrix_shared[12];
    __shared__ float currentTransform_shared[12];

    int tid = threadIdx.x;
    if(tid < 12) {
        invViewMatrix_shared[tid] = d_invViewMatrix[tid];
        currentTransform_shared[tid] = d_currentTransform[tid];
    }
    __syncthreads();

    float result[12] = {0.f};
    for (int i = 0; i < 3; ++i) {
        for (int j = 0; j < 4; ++j) {
            for (int k = 0; k < 4; ++k) {
                result[i * 4 + j] += invViewMatrix_shared[i * 4 + k] * currentTransform_shared[k * 4 + j];
            }
        }
    }
    
    if(tid < 12) {
        d_currentTransform[tid] = result[tid];
    }
}