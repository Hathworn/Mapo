#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimize the kernel for better performance and readability
__global__ void fastForwardDst(const int16_t* __restrict__ block, int16_t* __restrict__ coeff, int shift) 
{
    int i = threadIdx.x;
    int rnd_factor = 1 << (shift - 1);
    
    // Use registers for intermediate calculations
    int block0 = block[4 * i + 0];
    int block1 = block[4 * i + 1];
    int block2 = block[4 * i + 2];
    int block3 = block[4 * i + 3];

    int c0 = block0 + block3;
    int c1 = block1 + block3;
    int c2 = block0 - block1;
    int c3 = 74 * block2;

    // Compute coefficients using registers
    coeff[i] = (int16_t)((29 * c0 + 55 * c1 + c3 + rnd_factor) >> shift);
    coeff[4 + i] = (int16_t)((74 * (block0 + block1 - block3) + rnd_factor) >> shift);
    coeff[8 + i] = (int16_t)((29 * c2 + 55 * c0 - c3 + rnd_factor) >> shift);
    coeff[12 + i] = (int16_t)((55 * c2 - 29 * c1 + c3 + rnd_factor) >> shift);
}