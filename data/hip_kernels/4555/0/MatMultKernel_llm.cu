#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

#define BLOCK_SIZE 16  // Use a larger block size for better performance

// Global variable setup
const int radius = 3;
const int numSamples = 100;
const double learningRate = 0.15;
const int epochs = 1;
const int numNeurons = 20;
// Set up neurons
double inputLayer[2][numNeurons] = {0}; // Takes input and weights
double outputLayer[1][numNeurons] = {0}; // Takes weights and outputs
double *matrixA; // Temporarily initialized arrays for GPU memory allocation
double *matrixB;
double *matrixC;
double *matrixD;
double *matrixE;

// Calculates dot product of two arrays from a given pointer and returns a total - must be same size
__global__ void MatMultKernel(double *array1, double *array2, double *output, int arr1_rows, int arr1_cols, int arr2_cols) {
    double result = 0;
    __shared__ double subArray1[BLOCK_SIZE][BLOCK_SIZE]; // Use shared memory for tiles
    __shared__ double subArray2[BLOCK_SIZE][BLOCK_SIZE]; 

    int bIDx = blockIdx.x, bIDy = blockIdx.y;
    int tIDx = threadIdx.x, tIDy = threadIdx.y;
    
    int row = bIDy * BLOCK_SIZE + tIDy; // Global row index
    int col = bIDx * BLOCK_SIZE + tIDx; // Global column index
    
    for (int i = 0; i < (arr1_cols + BLOCK_SIZE - 1) / BLOCK_SIZE; ++i) { // Loop over tiles
        if (row < arr1_rows && i * BLOCK_SIZE + tIDx < arr1_cols) {
            subArray1[tIDy][tIDx] = array1[row * arr1_cols + i * BLOCK_SIZE + tIDx]; // Load tile into shared memory
        } else {
            subArray1[tIDy][tIDx] = 0; // Zero padding
        }
        
        if (col < arr2_cols && i * BLOCK_SIZE + tIDy < arr1_cols) {
            subArray2[tIDy][tIDx] = array2[(i * BLOCK_SIZE + tIDy) * arr2_cols + col]; // Load tile into shared memory
        } else {
            subArray2[tIDy][tIDx] = 0; // Zero padding
        }
        
        __syncthreads(); // Synchronize threads
        
        for (int j = 0; j < BLOCK_SIZE; ++j) {
            result += subArray1[tIDy][j] * subArray2[j][tIDx]; // Perform multiplication
        }
        
        __syncthreads(); // Ensure all threads are done
    }
    
    if (row < arr1_rows && col < arr2_cols) {
        output[row * arr2_cols + col] = result; // Output the result
    }
}