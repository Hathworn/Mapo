#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x; // Calculate global index
    if (i < N) { // Add boundary check to prevent out-of-bounds memory access
        c[i] = a[i] + b[i];
    }
}