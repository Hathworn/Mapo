#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void mergeLocation(const short2* __restrict__ loc_, float* __restrict__ x, float* __restrict__ y, const int npoints, float scale)
{
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    if (ptidx < npoints)
    {
        // Use __restrict__ to optimize memory access
        short2 loc = loc_[ptidx];

        // Directly calculate scaled values and store
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}