#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    if (i >= h || j >= w) return;

    int flow_row_offset  = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Pre-compute commonly used values
    float cx = u[flow_row_offset + j] * time_scale + (float)j + 1.0f;
    float cy = v[flow_row_offset + j] * time_scale + (float)i + 1.0f;

    float value = src[image_row_offset + j];
    
    float px, py, dx, dy, weight;
    int tx, ty;

    dx = modff(cx, &px);
    dy = modff(cy, &py);

    tx = (int)px;
    ty = (int)py;

    // Use loop unrolling for efficiency
    #pragma unroll
    for (int shift = 0; shift < 4; ++shift) {
        if (!((tx >= w) || (tx < 0) || (ty >= h) || (ty < 0))) {
            switch (shift) {
                case 0: weight = dx * dy; break; // Bottom-right corner
                case 1: weight = (1.0f - dx) * dy; tx -= 1; break; // Bottom-left corner
                case 2: weight = (1.0f - dx) * (1.0f - dy); ty -= 1; break; // Upper-left corner
                case 3: weight = dx * (1.0f - dy); tx += 1; break; // Upper-right corner
            }
            _atomicAdd(dst + ty * image_stride + tx, value * weight);
            _atomicAdd(normalization_factor + ty * image_stride + tx, weight);
        }

        if (shift == 2) {
            tx += 1; // Adjust tx for next iteration after upper-left corner adjustment
        }
    }
}