#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Calculate global linear index for the thread
    int globalIndex = blockIdx.y * blockDim.y * w + blockIdx.x * blockDim.x + threadIdx.y * w + threadIdx.x;
    
    // Calculate flow and image row offsets
    int flow_row_offset = (globalIndex / w) * flow_stride;
    int image_row_offset = (globalIndex / w) * image_stride;
    
    // Check bounds and load data efficiently
    if (globalIndex < w * h) {
        float u_ = u[flow_row_offset + (globalIndex % w)];
        float v_ = v[flow_row_offset + (globalIndex % w)];

        // Bottom left corner of target pixel
        float cx = u_ * time_scale + (float)(globalIndex % w) + 1.0f;
        float cy = v_ * time_scale + (float)(globalIndex / w) + 1.0f;

        // Pixel containing bottom left corner
        int tx = __float2int_rn(cx);
        int ty = __float2int_rn(cy);

        float value = src[image_row_offset + (globalIndex % w)];
        
        // Fill pixel within bounds efficiently
        if (tx >= 0 && tx < w && ty >= 0 && ty < h) {
            atomicAdd(dst + ty * image_stride + tx, value);
        }
    }
}