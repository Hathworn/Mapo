#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void NormalizeKernel(const float *normalization_factor, int w, int h, int s, float *image)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    // Optimize: Use shared memory for normalization factor
    __shared__ float sharedNormalizationFactor[BLOCKDIM_X * BLOCKDIM_Y];

    int localIdx = threadIdx.y * blockDim.x + threadIdx.x;

    if (i < h && j < w) {
        const int pos = i * s + j;
        
        // Load normalization factor to shared memory
        sharedNormalizationFactor[localIdx] = normalization_factor[pos];
        __syncthreads();
        
        float scale = sharedNormalizationFactor[localIdx];

        // Prevent division by zero
        float invScale = (scale == 0.0f) ? 1.0f : (1.0f / scale);

        // Normalize image using preloaded factor from shared memory
        image[pos] *= invScale;
    }
}