#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void vecMult(float* a, float* b, float* c, const int N) {
    // Calculate the global index of the thread
    const int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * gridDim.x * blockDim.x;

    // Ensure index is within bounds and perform multiplication
    if (i < N)
        c[i] = a[i] * b[i];
}