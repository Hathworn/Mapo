#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void rowDiv(float* a, float* b, float* c, int M, int N) {
    // Calculate global index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure index is within bounds
    if (i < M * N) {
        // Divide a[i] by the corresponding row value in b
        c[i] = a[i] / b[blockIdx.x];
    }
}