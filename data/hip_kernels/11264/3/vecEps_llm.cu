#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Use shared memory to reduce global memory access count
__global__ void vecEps(float* a, const int N) {
    const int i = gridDim.x * blockDim.x * blockIdx.y + blockIdx.x * blockDim.x + threadIdx.x;
    
    if (i < N) {
        float val = a[i];
        if (val < EPS) {
            a[i] = EPS;
        }
    }
}