#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void vecDiv(float* a, float* b, float* c, const int N) {
    // Calculate global thread index
    const int i = blockIdx.x * blockDim.x + threadIdx.x + blockIdx.y * blockDim.x * gridDim.x;
    
    // Perform division if within bounds
    if(i < N) { 
        c[i] = a[i] / b[i]; 
        // c[i] = __fdividef(a[i], b[i]);  // Uncomment for faster, less-accurate divide
    }
}