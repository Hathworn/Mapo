#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void colMul(float* a, float* b, float* c, int M, int N) {
    // Calculate global row index
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    // Ensure thread is within bounds
    if (row < M) {
        // Precompute column offset
        int colOffset = blockIdx.y * M;
        // Compute global element index
        int globalIndex = row + colOffset;
        // Perform multiplication
        c[globalIndex] = a[globalIndex] * b[row];
    }
}