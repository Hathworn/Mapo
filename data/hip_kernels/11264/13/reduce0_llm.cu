#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void reduce0(float *g_idata, float *g_odata, int N) {
    extern __shared__ float sdata[];

    int tid = threadIdx.x;
    int i = blockIdx.x * (blockDim.x * 2) + threadIdx.x;
    
    // Load elements into shared memory; handle boundary
    float mySum = (i < N) ? g_idata[i] : 0.0f;
    if (i + blockDim.x < N) mySum += g_idata[i + blockDim.x];
    sdata[tid] = mySum;

    __syncthreads();

    // Unroll loop for faster reduction
    for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Warp-level reduction
    if (tid < 32) {
        volatile float* smem = sdata;
        smem[tid] += smem[tid + 32];
        smem[tid] += smem[tid + 16];
        smem[tid] += smem[tid + 8];
        smem[tid] += smem[tid + 4];
        smem[tid] += smem[tid + 2];
        smem[tid] += smem[tid + 1];
    }

    // Write the result for this block to global memory
    if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}