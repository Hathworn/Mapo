#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void colDiv(float* a, float* b, float* c, int M, int N) {
    // Calculate the global thread index
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Convert 2D grid location into a column index
    int j = blockIdx.y; 
    
    // Ensure the index is within bounds
    if (i < M && j < N) {
        // Calculate linear index for 2D array
        int ind = i + j * M;
        
        // Perform division operation
        c[ind] = a[ind] / b[i];
    }
}