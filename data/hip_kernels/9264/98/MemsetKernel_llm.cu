#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void MemsetKernel(const float value, int w, int h, float *image)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y; // Calculate global row index
    int j = blockIdx.x * blockDim.x + threadIdx.x; // Calculate global column index
    
    if (i < h && j < w) { // Ensure within bounds
        const int pos = i * w + j; // Compute linear index
        image[pos] = value; // Assign value
    }
}