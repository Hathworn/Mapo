#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void mergeLocation(const short2* loc_, float* x, float* y, const int npoints, float scale)
{
    // Calculate global thread ID
    const int ptidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Check bounds
    if (ptidx < npoints)
    {
        // Use registers for local variables
        short2 loc = loc_[ptidx];

        // Perform scaled assignment
        x[ptidx] = loc.x * scale;
        y[ptidx] = loc.y * scale;
    }
}