#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF1x1(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *dst)
{
    // Calculate global thread indices for i and j
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    int i = threadIdx.y + blockDim.y * blockIdx.y;

    // Return if the indices are out of bounds
    if (i >= h || j >= w) return;

    int flow_row_offset = i * flow_stride;
    int image_row_offset = i * image_stride;

    // Load u and v components
    float u_ = u[flow_row_offset + j];
    float v_ = v[flow_row_offset + j];

    // Calculate target pixel's bottom left corner (cx, cy)
    float cx = u_ * time_scale + (float)j + 1.0f;
    float cy = v_ * time_scale + (float)i + 1.0f;

    // Convert float coordinates to integer pixel indices
    int tx = __float2int_rn(cx);
    int ty = __float2int_rn(cy);

    // Load the value from the source
    float value = src[image_row_offset + j];

    // Check if target pixel indices are within bounds and atomically add value
    if (tx >= 0 && tx < w && ty >= 0 && ty < h)
    {
        atomicAdd(dst + ty * image_stride + tx, value);
    }
}