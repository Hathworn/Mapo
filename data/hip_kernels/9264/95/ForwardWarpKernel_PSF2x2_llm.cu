#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"
__device__ __forceinline__ float imag(const float2& val)
{
    return val.y;
}

__global__ void ForwardWarpKernel_PSF2x2(const float *u, const float *v, const float *src, const int w, const int h, const int flow_stride, const int image_stride, const float time_scale, float *normalization_factor, float *dst)
{
    // Calculate 1D block index for better memory coalescing
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int idy = blockDim.y * blockIdx.y + threadIdx.y;

    if (idx >= w || idy >= h) return;

    int flow_row_offset  = idy * flow_stride;
    int image_row_offset = idy * image_stride;

    // Bottom left corner of a target pixel
    float cx = u[flow_row_offset + idx] * time_scale + (float)idx + 1.0f;
    float cy = v[flow_row_offset + idx] * time_scale + (float)idy + 1.0f;

    // Calculate integer coords and sub-pixel offsets
    float px, py;
    float dx = modff(cx, &px);
    float dy = modff(cy, &py);

    int tx = (int)px;
    int ty = (int)py;
    float value = src[image_row_offset + idx];

    // Precompute and reuse common weights
    float w_dx_dy = dx * dy;
    float w_one_minus_dx_dy = (1.0f - dx) * dy;
    float w_dx_one_minus_dy = dx * (1.0f - dy);
    float w_one_minus_dx_one_minus_dy = (1.0f - dx) * (1.0f - dy);

    // Atomic adds to ensure race conditions are avoided
    if (tx < w && tx >= 0 && ty < h && ty >= 0)
    {
        atomicAdd(dst + ty * image_stride + tx, value * w_dx_dy);
        atomicAdd(normalization_factor + ty * image_stride + tx, w_dx_dy);
    }

    if (tx - 1 < w && tx - 1 >= 0 && ty < h && ty >= 0)
    {
        atomicAdd(dst + ty * image_stride + (tx - 1), value * w_one_minus_dx_dy);
        atomicAdd(normalization_factor + ty * image_stride + (tx - 1), w_one_minus_dx_dy);
    }

    if (tx - 1 < w && tx - 1 >= 0 && ty - 1 < h && ty - 1 >= 0)
    {
        atomicAdd(dst + (ty - 1) * image_stride + (tx - 1), value * w_one_minus_dx_one_minus_dy);
        atomicAdd(normalization_factor + (ty - 1) * image_stride + (tx - 1), w_one_minus_dx_one_minus_dy);
    }

    if (tx < w && tx >= 0 && ty - 1 < h && ty - 1 >= 0)
    {
        atomicAdd(dst + (ty - 1) * image_stride + tx, value * w_dx_one_minus_dy);
        atomicAdd(normalization_factor + (ty - 1) * image_stride + tx, w_dx_one_minus_dy);
    }
}
```
