#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void test_launch_cuda_native(float * scalar, float * vector, int sxy, int sx , int sy , int sz , int stride)
{
    int id_x = threadIdx.x + blockIdx.x * blockDim.x;
    int id_y = threadIdx.y + blockIdx.y * blockDim.y;
    int id_z = threadIdx.z + blockIdx.z * blockDim.z;

    if (id_x >= sx || id_y >= sy || id_z >= sz) return; // Combine bounds check

    int idx = id_z * sxy + id_y * sx + id_x; // Compute linear index once for reuse

    float s = scalar[idx];

    float v[3];
    v[0] = vector[idx + 0 * stride];
    v[1] = vector[idx + 1 * stride];
    v[2] = vector[idx + 2 * stride];

    printf("Grid point from CUDA %d %d %d     scalar: %f  vector: %f %f %f \n", id_x, id_y, id_z, s, v[0], v[1], v[2]);
}