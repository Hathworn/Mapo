#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_count(float *prp_0, int sz)
{
    // Calculate linear index for 3D grid
    int tid = blockIdx.z * blockDim.z * sz * sz + 
              blockIdx.y * blockDim.y * sz + 
              blockIdx.x * blockDim.x + 
              threadIdx.z * sz * sz + 
              threadIdx.y * sz + 
              threadIdx.x;

    if (tid < sz * sz * sz) // Ensure write is within bounds
    {
        prp_0[tid] = tid; // Assign the value directly using the calculated index
    }
}