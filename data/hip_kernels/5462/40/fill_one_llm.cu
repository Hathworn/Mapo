#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void fill_one(float * prp_0, int sz)
{
    // Calculate linear thread index for 3D grid
    int idx = blockIdx.z * blockDim.z * sz * sz + blockIdx.y * blockDim.y * sz + blockIdx.x * blockDim.x;
    idx += threadIdx.z * sz * sz + threadIdx.y * sz + threadIdx.x;

    // Check if index is within bounds before assignment
    if (idx < sz * sz * sz) {
        prp_0[idx] = 1.0f;
    }
}