#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void sync_conv_groups() { 
    // Ensure all threads in the block reach this point before proceeding.
    __syncthreads(); 
}