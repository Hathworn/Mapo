#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void relu_gpu_forward(float *out, float *in, int64_t N) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // Use loop unrolling for better performance
    for (int i = tid; i < N; i += blockDim.x * gridDim.x) {
        out[i] = in[i] > 0 ? in[i] : 0;
    }
}