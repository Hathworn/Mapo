#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Load accuracy into a register to minimize memory accesses
    float acc = *accuracy;
    // Perform the division
    acc /= N;
    // Store the result back to global memory
    *accuracy = acc;
}