#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

// Optimized initialization of hiprand state for each thread
__global__ void kSetupCurand(hiprandState *state, unsigned long long seed) {
    // Calculate unique thread index across all blocks
    const uint tidx = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize each thread with the hiprand seed and sequence number
    hiprand_init(seed, tidx, 0, &state[tidx]);
}