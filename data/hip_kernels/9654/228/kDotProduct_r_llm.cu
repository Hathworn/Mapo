#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kDotProduct_r(float* a, float* b, float* target, const uint numElements) {
    __shared__ float shmem[DP_BLOCKSIZE];
    
    uint eidx = DP_BLOCKSIZE * blockIdx.x + threadIdx.x;
    float sum = 0.0f; // Accumulate in a register
    while (eidx < numElements) {
        sum += a[eidx] * b[eidx];
        eidx += gridDim.x * DP_BLOCKSIZE;
    }
    shmem[threadIdx.x] = sum;
    __syncthreads();
    
    // Use a loop to perform reduction instead of unrolling manually
    for (uint stride = DP_BLOCKSIZE / 2; stride > 0; stride /= 2) {
        if (threadIdx.x < stride) {
            shmem[threadIdx.x] += shmem[threadIdx.x + stride];
        }
        __syncthreads();
    }
    
    // Write result from the first thread in the block
    if (threadIdx.x == 0) {
        target[blockIdx.x] = shmem[0];
    }
}