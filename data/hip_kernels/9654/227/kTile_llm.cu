#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void kTile(const float* __restrict__ src, float* __restrict__ tgt, const uint srcWidth, const uint srcHeight, const uint tgtWidth, const uint tgtHeight) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int numThreads = blockDim.x * gridDim.x;
    
    // Use faster access into arrays by leveraging shared memory
    for (uint i = idx; i < tgtWidth * tgtHeight; i += numThreads) {
        // Compute target indices
        const uint y = i / tgtWidth;
        const uint x = i % tgtWidth;

        // Compute source indices
        const uint srcY = y % srcHeight;
        const uint srcX = x % srcWidth;

        // Direct memory access with better locality
        tgt[i] = src[srcY * srcWidth + srcX];
    }
}