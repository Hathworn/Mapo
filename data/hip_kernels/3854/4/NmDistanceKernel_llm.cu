#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void NmDistanceKernel(int b, int n, const float *xyz, int m, const float *xyz2, float *result, int *result_i) {
    const int batch = 512;
    __shared__ float buf[batch * 3];

    for (int i = blockIdx.x; i < b; i += gridDim.x) {
        for (int k2 = 0; k2 < m; k2 += batch) {
            int end_k = min(m, k2 + batch) - k2;

            // Load block elements into shared memory
            for (int j = threadIdx.x; j < end_k * 3; j += blockDim.x) {
                buf[j] = xyz2[(i * m + k2) * 3 + j];
            }
            __syncthreads();

            for (int j = threadIdx.x + blockIdx.y * blockDim.x; j < n; j += blockDim.x * gridDim.y) {
                float x1 = xyz[(i * n + j) * 3 + 0];
                float y1 = xyz[(i * n + j) * 3 + 1];
                float z1 = xyz[(i * n + j) * 3 + 2];
                int best_i = 0;
                float best = FLT_MAX;  // Initialize with maximum float value

                // Unroll the loop by 4
                int end_ka = end_k - (end_k & 3);
                for (int k = 0; k < end_ka; k += 4) {
                    #pragma unroll
                    for (int offset = 0; offset < 4; offset++) {
                        float x2 = buf[(k + offset) * 3 + 0] - x1;
                        float y2 = buf[(k + offset) * 3 + 1] - y1;
                        float z2 = buf[(k + offset) * 3 + 2] - z1;
                        float d = x2 * x2 + y2 * y2 + z2 * z2;
                        if (d < best) {  // Check if current distance is a new best
                            best = d;
                            best_i = k + k2 + offset;
                        }
                    }
                }

                // Handle the remainder of the elements
                for (int k = end_ka; k < end_k; k++) {
                    float x2 = buf[k * 3 + 0] - x1;
                    float y2 = buf[k * 3 + 1] - y1;
                    float z2 = buf[k * 3 + 2] - z1;
                    float d = x2 * x2 + y2 * y2 + z2 * z2;
                    if (d < best) {
                        best = d;
                        best_i = k + k2;
                    }
                }

                // Update result if the current best is better
                if (k2 == 0 || result[(i * n + j)] > best) {
                    result[(i * n + j)] = best;
                    result_i[(i * n + j)] = best_i;
                }
            }
            __syncthreads();
        }
    }
}