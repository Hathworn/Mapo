#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void NmDistanceGradKernel(int b, int n, const float *xyz1, int m, const float *xyz2, const float *grad_dist1, const int *idx1, float *grad_xyz1, float *grad_xyz2) {
    // Calculate block and thread indices for accessing data
    int batch_index = blockIdx.x;
    int point_index = threadIdx.x + blockIdx.y * blockDim.x;

    // Parallelize processing for all batches and points
    if (batch_index < b && point_index < n) {
        // Use registers to optimize memory access
        float x1 = xyz1[(batch_index * n + point_index) * 3 + 0];
        float y1 = xyz1[(batch_index * n + point_index) * 3 + 1];
        float z1 = xyz1[(batch_index * n + point_index) * 3 + 2];
        int j2 = idx1[batch_index * n + point_index];
        float x2 = xyz2[(batch_index * m + j2) * 3 + 0];
        float y2 = xyz2[(batch_index * m + j2) * 3 + 1];
        float z2 = xyz2[(batch_index * m + j2) * 3 + 2];
        float g = grad_dist1[batch_index * n + point_index] * 2;

        // Minimize atomic operations for faster performance
        float delta_x = g * (x1 - x2);
        float delta_y = g * (y1 - y2);
        float delta_z = g * (z1 - z2);

        atomicAdd(&(grad_xyz1[(batch_index * n + point_index) * 3 + 0]), delta_x);
        atomicAdd(&(grad_xyz1[(batch_index * n + point_index) * 3 + 1]), delta_y);
        atomicAdd(&(grad_xyz1[(batch_index * n + point_index) * 3 + 2]), delta_z);
        atomicAdd(&(grad_xyz2[(batch_index * m + j2) * 3 + 0]), -delta_x);
        atomicAdd(&(grad_xyz2[(batch_index * m + j2) * 3 + 1]), -delta_y);
        atomicAdd(&(grad_xyz2[(batch_index * m + j2) * 3 + 2]), -delta_z);
    }
}