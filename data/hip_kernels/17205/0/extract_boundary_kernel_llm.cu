#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;
#define ITERATIONS 40000

enum pixel_position {INSIDE_MASK, BOUNDRY, OUTSIDE};

__global__ void extract_boundary_kernel(float *maskIn, int *boundryPixelArray, int source_nchannel, int source_width, int source_height){
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    // Check if thread is within image bounds
    if (x < source_width && y < source_height) {
        for (int channel = 0; channel < source_nchannel; channel++) {
            int id = x + source_width * y + source_width * source_height * channel;

            // Directly mark corners and edges as OUTSIDE
            if ((x == 0 || x == source_width - 1 || y == 0 || y == source_height - 1) && maskIn[id]) {
                boundryPixelArray[id] = OUTSIDE;
            } else {
                // Calculate neighboring indices once
                int id_right = x + 1 + y * source_width + channel * source_width * source_height;
                int id_left = x - 1 + y * source_width + channel * source_width * source_height;
                int id_up = x + (y + 1) * source_width + channel * source_width * source_height;
                int id_down = x + (y - 1) * source_width + channel * source_width * source_height;

                // Flags whether the current pixel is inside the mask boundary
                bool is_in_mask = maskIn[id] >= 0.5;
                bool neighbors_in_mask = maskIn[id_right] >= 0.5 && maskIn[id_left] >= 0.5 
                                         && maskIn[id_up] >= 0.5 && maskIn[id_down] >= 0.5;

                // Set boundary pixel array based on mask condition
                if (is_in_mask && neighbors_in_mask) {
                    boundryPixelArray[id] = INSIDE_MASK;
                } else if (is_in_mask) {
                    boundryPixelArray[id] = BOUNDRY;
                } else {
                    boundryPixelArray[id] = OUTSIDE;
                }
            }
        }
    }
}