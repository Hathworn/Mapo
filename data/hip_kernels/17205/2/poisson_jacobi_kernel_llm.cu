#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;
#define ITERATIONS 40000

enum pixel_position {INSIDE_MASK, BOUNDRY, OUTSIDE};

__global__ void poisson_jacobi_kernel(float *targetimg, float *outimg, int *boundary_array, int c, int w, int h, int boundBoxMinX, int boundBoxMaxX, int boundBoxMinY, int boundBoxMaxY) {
    int x = threadIdx.x + blockIdx.x * blockDim.x + boundBoxMinX;
    int y = threadIdx.y + blockIdx.y * blockDim.y + boundBoxMinY;

    // Check if x, y are within valid range before processing
    if (x < boundBoxMinX || x >= boundBoxMaxX || y < boundBoxMinY || y >= boundBoxMaxY)
        return;

    for (int channel = 0; channel < c; channel++) {
        int id = x + y * w + channel * w * h;

        // Avoid out-of-bounds accesses using boundary conditions
        int idx_nextX = (x + 1 < w) ? (x + 1 + y * w + channel * w * h) : id;
        int idx_prevX = (x - 1 >= 0) ? (x - 1 + y * w + channel * w * h) : id;
        int idx_nextY = (y + 1 < h) ? (x + (y + 1) * w + channel * w * h) : id;
        int idx_prevY = (y - 1 >= 0) ? (x + (y - 1) * w + channel * w * h) : id;

        if (boundary_array[id] == INSIDE_MASK) {
            double neighbor_target = targetimg[idx_nextY] + targetimg[idx_nextX] + targetimg[idx_prevX] + targetimg[idx_prevY];
            double neighbor_output = outimg[idx_nextY] + outimg[idx_nextX] + outimg[idx_prevX] + outimg[idx_prevY];
            outimg[id] = 0.25f * (4 * targetimg[id] - neighbor_target + neighbor_output); // Use 'f' for float operations
        }
    }
}