#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
using namespace std;
#define ITERATIONS 40000

enum pixel_position {INSIDE_MASK, BOUNDRY, OUTSIDE};

__global__ void merge_without_blend_kernel(float *srcimg, float *targetimg, float *outimg, int *boundary_array, int source_nchannel, int source_width, int source_height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (x < source_width && y < source_height) { // Ensure within image bounds
        int base_id = x + y * source_width;
        for (int channel = 0; channel < source_nchannel; channel++) {
            int id = base_id + channel * source_width * source_height;
            // Use ternary operator for compact conditional assignment
            outimg[id] = (boundary_array[id] == INSIDE_MASK) ? targetimg[id] : srcimg[id];
        }
    }
}