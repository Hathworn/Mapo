#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void AccuracyDivideKernel(const int N, float* accuracy) {
    // Use the first thread in block 0 to perform the division
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        *accuracy /= N;
    }
}