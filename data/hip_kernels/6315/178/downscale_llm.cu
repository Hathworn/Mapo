#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ int translate_idx_inv(int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    x = ii / d1;
    w = w * scale_factor + off_x;
    z = z * scale_factor + off_y;
    d2 *= scale_factor;
    d3 *= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
    int x, y, z, w;
    w = ii % d3;
    ii = ii / d3;
    z = ii % d2;
    ii = ii / d2;
    y = ii % d1;
    x = ii / d1;
    w /= scale_factor;
    z /= scale_factor;
    d2 /= scale_factor;
    d3 /= scale_factor;
    return (((x * d1 + y) * d2) + z) * d3 + w;
}

__device__ __forceinline__ size_t idx(const size_t nc, const size_t height, const size_t width, const size_t y, const size_t x) {
    return (nc * height + y) * width + x;
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data, long no_elements, int scale_factor, int d1, int d2, int d3) {
    long ii = threadIdx.x + blockDim.x * blockIdx.x + threadIdx.y * blockDim.y * gridDim.x * blockDim.x * blockIdx.y;
    if (ii >= no_elements) return;
    
    const int ipidx_base = translate_idx_inv(ii, d1, d2, d3, scale_factor, 0, 0); // Pre-compute base index
    for (int i = 0; i < scale_factor * scale_factor; i++) {
        int offset_x = i % scale_factor;
        int offset_y = i / scale_factor;
        int ipidx = ipidx_base + offset_x + offset_y * d3 * scale_factor; // Compute each ipidx with offsets
        gradInput_data[ii] += gradOutput_data[ipidx];
    }
}