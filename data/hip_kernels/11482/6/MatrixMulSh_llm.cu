#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMulSh( float *Md , float *Nd , float *Pd , const int WIDTH )
{
    // Utilizing shared memory to load matrix tiles
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // Calculate thread id
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

    // Initialize the element in the result matrix to zero
    float Pvalue = 0.0f;

    for (int m = 0; m < WIDTH / TILE_WIDTH; m++) // m indicates number of phases
    {
        // Load tiles into shared memory
        Mds[threadIdx.y][threadIdx.x] = Md[row * WIDTH + (m * TILE_WIDTH + threadIdx.x)];
        Nds[threadIdx.y][threadIdx.x] = Nd[(m * TILE_WIDTH + threadIdx.y) * WIDTH + col];
        __syncthreads(); // Ensure all threads have loaded their data

        // Compute the partial result for this tile
        for (int k = 0; k < TILE_WIDTH; k++)
        {
            Pvalue += Mds[threadIdx.y][k] * Nds[k][threadIdx.x];
        }
        __syncthreads(); // Synchronize before loading new tiles
    }

    // Write the computed value to the output matrix
    Pd[row * WIDTH + col] = Pvalue;
}