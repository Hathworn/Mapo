#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"
__global__ void gaxpy(double *y, double *a, double *x, int m, int n) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];
    
    // Load and calculate product directly in shared memory
    if (bid < m && tid < n) {
        dots_s[tid] = a[bid * n + tid] * x[tid];
    }
    __syncthreads();
    
    // Use a single thread to perform reduction using shared memory
    if (tid == 0) {
        double sum = 0.0;
        for (int i = 0; i < n; i++) {
            sum += dots_s[i];
        }
        y[bid] = sum;
    }
}