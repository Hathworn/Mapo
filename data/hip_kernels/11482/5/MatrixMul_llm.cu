#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void MatrixMul(float *Md, float *Nd, float *Pd, const int WIDTH) {
    // Calculate thread id
    unsigned int col = TILE_WIDTH * blockIdx.x + threadIdx.x;
    unsigned int row = TILE_WIDTH * blockIdx.y + threadIdx.y;

    // Initialize accumulator to avoid repeated indexing in Pd
    float Pvalue = 0;

    // Use a single unrolled loop if possible for better performance
    for (int k = 0; k < WIDTH; k++) {
        Pvalue += Md[row * WIDTH + k] * Nd[k * WIDTH + col];
    }

    // Write computed value once at the end
    Pd[row * WIDTH + col] = Pvalue;
}