#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void multMatriz(float *da, float *db, float *dc, int num) {
    // Calculate global row index
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    // Calculate global column index
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    
    // Ensure within bounds
    if (row < num && col < num) {
        // Initialize sum
        float sum = 0;
        // Calculate dot product for one element of the result matrix
        for (int k = 0; k < num; ++k) {
            sum += da[row * num + k] * db[k * num + col];
        }
        // Assign the computed value to the result matrix
        dc[row * num + col] = sum;
    }
}