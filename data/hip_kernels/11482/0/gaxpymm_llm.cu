#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void gaxpymm(double *y, double *a, double *b, int m, int n, int p) {
    int bid = blockIdx.x;
    int tid = threadIdx.x;
    extern __shared__ double dots_s[];
    
    if (bid < m) {
        if (tid < n) {
            // Compute partial dot products
            for (int c = 0; c < p; c++) {
                dots_s[tid * p + c] = a[bid * n + tid] * b[tid * p + c];
            }
        }
        
        __syncthreads();
        
        if (tid == 0) {
            // Aggregate the results
            for (int c = 0; c < p; c++) {
                double sum = 0.0;
                for (int i = 0; i < n; i++) {
                    sum += dots_s[i * p + c];
                }
                y[bid * p + c] = sum;
            }
        }
    }
}