#include "hip/hip_runtime.h"
```c
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void suma(int a, int b, int *c) {
    // Use shared memory if possible to store intermediates
    // Calculate thread index for efficiency
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // Perform operation based on calculated index
    if (index == 0) {
        *c = a + b;
    }
}