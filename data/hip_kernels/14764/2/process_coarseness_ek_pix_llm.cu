#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__global__ void process_coarseness_ek_pix(double *output_ak, double *output_ekh, double *output_ekv, int colsize, int rowsize, long lenOf_ek)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x; // Correct indices for 2D grid
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    if (x < rowsize && y < colsize) // Ensure within bounds
    {
        int posx1 = x + lenOf_ek; // Pre-calculate index positions for x
        int posx2 = x - lenOf_ek;
        int posy1 = y + lenOf_ek; // Pre-calculate index positions for y
        int posy2 = y - lenOf_ek;

        if (posx1 < rowsize && posx2 >= 0)
        {
            double input1 = output_ak[y * rowsize + posx1];
            double input2 = output_ak[y * rowsize + posx2];
            output_ekh[y * rowsize + x] = fabs(input1 - input2);
        }
        else 
        {
            output_ekh[y * rowsize + x] = 0;
        }

        if (posy1 < colsize && posy2 >= 0)
        {
            double input1 = output_ak[posy1 * rowsize + x];
            double input2 = output_ak[posy2 * rowsize + x];
            output_ekv[y * rowsize + x] = fabs(input1 - input2);
        }
        else 
        {
            output_ekv[y * rowsize + x] = 0;
        }
    }
}