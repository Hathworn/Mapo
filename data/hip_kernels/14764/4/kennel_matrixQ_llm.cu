#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double atomicAdd_dB(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));
        // Integer comparison avoids hang if NaN
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__ double atomicAdd_d(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                                             __longlong_as_double(assumed)));
        // Integer comparison avoids hang if NaN
    } while (assumed != old);

    return __longlong_as_double(old);
}

__global__ void kennel_matrixQ(double *d_P, double *d_px, double *d_py, double *d_Q, int Ng)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= Ng * Ng) return;  // Ensure tid is within bounds

    int rownum = tid / Ng;
    int colnum = tid % Ng;
    double sum = 0.0;  // Accumulate partial results

    for (int k = 0; k < Ng; ++k)
    {
        double px = d_px[rownum];
        double py = d_py[k];
        if (px != 0 && py != 0)  // Protect against division by zero
        {
            double temp = d_P[k + rownum * Ng] * d_P[k + colnum * Ng] / px / py;
            sum += temp;  // Accumulate results
        }
    }
    atomicAdd_dB(&d_Q[(tid + Ng)], sum); // Use atomic add for final result
}