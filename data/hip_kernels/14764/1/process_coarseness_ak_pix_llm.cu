#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ double efficientLocalMean_dev(const long x, const long y, const long k, double *input_img, int rowsize, int colsize) {
    long k2 = k / 2;

    long dimx = rowsize;
    long dimy = colsize;

    // Calculate region of interest
    long starty = max(0, y - k2);
    long startx = max(0, x - k2);
    long stopy = min(dimy - 1, y + k2 - 1);
    long stopx = min(dimx - 1, x + k2 - 1);

    // Fetch boundary values ensuring no out-of-bound memory access
    double links = (startx - 1 < 0) ? 0 : *(input_img + (stopy * dimx + startx - 1));
    double oben = (starty - 1 < 0) ? 0 : *(input_img + ((stopy - 1) * dimx + startx));
    double obenlinks = ((starty - 1 < 0) || (startx - 1 < 0)) ? 0 : *(input_img + ((stopy - 1) * dimx + startx - 1));
    double unten = *(input_img + (stopy * dimx + startx));

    long counter = (stopy - starty + 1) * (stopx - startx + 1);
    return (unten - links - oben + obenlinks) / counter;
}

__global__ void process_coarseness_ak_pix(double *output_ak, double *input_img, int colsize, int rowsize, long lenOf_ak) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Ensure within image boundaries
    if (y < colsize && x < rowsize) {
        int index = y * rowsize + x;
        output_ak[index] = efficientLocalMean_dev(x, y, lenOf_ak, input_img, rowsize, colsize);
    }
}