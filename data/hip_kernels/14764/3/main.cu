#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdio.h>
#include <string.h>
#include <getopt.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include "kernel_conv.cu"
#include <chrono>
#include <iostream>
using namespace std;
using namespace std::chrono;
int main(int argc, char **argv) {
hipSetDevice(0);

    int XSIZE = 512; 
    int YSIZE = 512;
    int BLOCKX = 16;
    int BLOCKY = 16;

    if (argc > 1) XSIZE = atoi(argv[1]);
    if (argc > 2) YSIZE = atoi(argv[2]);
    if (argc > 3) BLOCKX = atoi(argv[3]);
    if (argc > 4) BLOCKY = atoi(argv[4]);
    double2 *d_c = NULL;
hipMalloc(&d_c, XSIZE*YSIZE);
double *d_a = NULL;
hipMalloc(&d_a, XSIZE*YSIZE);
double2 *d_b = NULL;
hipMalloc(&d_b, XSIZE*YSIZE);
int *d_row_vect = NULL;
hipMalloc(&d_row_vect, XSIZE*YSIZE);
int *d_col_vect = NULL;
hipMalloc(&d_col_vect, XSIZE*YSIZE);
int ma = 1;
int na = 1;
int mb = 1;
int nb = 1;
int mc = 1;
int nc = 1;
int iXSIZE= XSIZE;
int iYSIZE= YSIZE;
while(iXSIZE%BLOCKX!=0) {
iXSIZE++;
}
while(iYSIZE%BLOCKY!=0) {
iYSIZE++;
}
dim3 gridBlock(iXSIZE/BLOCKX, iYSIZE/BLOCKY);
dim3 threadBlock(BLOCKX, BLOCKY);
hipFree(0);
kernel_conv<<<gridBlock, threadBlock>>>(d_c,d_a,d_b,d_row_vect,d_col_vect,ma,na,mb,nb,mc,nc);
hipDeviceSynchronize();
for (int loop_counter = 0; loop_counter < 5; ++loop_counter) {
kernel_conv<<<gridBlock, threadBlock>>>(d_c,d_a,d_b,d_row_vect,d_col_vect,ma,na,mb,nb,mc,nc);
}
hipDeviceSynchronize();
auto start = steady_clock::now();
for (int loop_counter = 0; loop_counter < 5; loop_counter++) {
kernel_conv<<<gridBlock, threadBlock>>>(d_c,d_a,d_b,d_row_vect,d_col_vect,ma,na,mb,nb,mc,nc);
}
hipDeviceSynchronize();
auto end = steady_clock::now();
auto usecs = duration_cast<duration<float, microseconds::period>>(end - start);
cout << '[' << usecs.count() << ',' << '(' << BLOCKX << ',' << BLOCKY << ')' << ',' << '(' << XSIZE << ',' << YSIZE << ')' << ']' << endl;
}