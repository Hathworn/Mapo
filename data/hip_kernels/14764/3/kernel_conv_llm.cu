#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "includes.h"

__device__ void compute_conv(int row, int col, double2 *d_c, double *d_a, double2 *d_b, int *o_row_vect, int *o_col_vect, int ma, int na, int mb, int nb, int mc, int nc) {
    int count_row = o_row_vect[row];
    int count_col = o_col_vect[col];
    int k_row_idx = max(row - (ma - 1), 0); // Simplified calculation using max()
    int k_col_start_idx = max(col - (na - 1), 0); // Simplified calculation using max()
    int k_col_idx = k_col_start_idx;
    int i_row_idx = min(row, ma - 1); // Simplified calculation using min()
    int i_col_idx = min(col, na - 1); // Simplified calculation using min()
    int i_col_start_idx = i_col_idx;

    for (int row_idx = 0; row_idx < count_row; row_idx++) {
        for (int col_idx = 0; col_idx < count_col; col_idx++) {
            double a_val = d_a[i_col_idx + na * i_row_idx]; // Cache the value of d_a
            double2 b_val = d_b[k_col_idx + nb * k_row_idx]; // Cache the value of d_b
            d_c[col + nc * row].x += a_val * b_val.x; // Optimize repeated array access
            d_c[col + nc * row].y += a_val * b_val.y; // Optimize repeated array access

            k_col_idx++;
            i_col_idx--;
        }
        k_row_idx++;
        i_row_idx--;
        k_col_idx = k_col_start_idx;
        i_col_idx = i_col_start_idx;
    }
}

__global__ void kernel_conv(double2 *d_c, double *d_a, double2 *d_b, int *d_row_vect, int *d_col_vect, int ma, int na, int mb, int nb, int mc, int nc) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int num_threads = gridDim.x * blockDim.x;

    for (int i = idx; i < (mc * nc); i += num_threads) {
        int rownum = i / nc;
        int colnum = i % nc;
        compute_conv(rownum, colnum, d_c, d_a, d_b, d_row_vect, d_col_vect, ma, na, mb, nb, mc, nc);
    }
}